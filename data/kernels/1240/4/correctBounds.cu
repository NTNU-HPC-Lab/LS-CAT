#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void correctBounds(double *d_ub, double *d_lb, int nRxns, double *d_prevPoint, double alpha, double beta, double *d_centerPoint, double *points, int pointsPerFile, int pointCount, int index){
int newindex = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for(int i=newindex;i<nRxns ;i+=stride){
if(points[pointCount+pointsPerFile*i]>d_ub[i]){
points[pointCount+pointsPerFile*i]=d_ub[i];
}else if(points[pointCount+pointsPerFile*i]<d_lb[i]){
points[pointCount+pointsPerFile*i]=d_lb[i];
}
d_prevPoint[nRxns*index+i]=points[pointCount+pointsPerFile*i];
d_centerPoint[nRxns*index+i]=alpha*d_centerPoint[nRxns*index+i]+beta*points[pointCount+pointsPerFile*i];
}
}