#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reprojectPoint(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount, int index){
int newindex = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for(int i=newindex;i<nRxns-istart;i+=stride){
d_umat[nRxns*index+i]=0;//d_umat now is d_tmp
for(int j=0;j<nRxns;j++){
d_umat[nRxns*index+i]+=d_N[j+i*nRxns]*points[pointCount+pointsPerFile*j];//here t(N)*Pt
}
}
}