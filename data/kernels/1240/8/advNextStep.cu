#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advNextStep(double *d_prevPoint, double *d_umat, double d_stepDist, int nRxns, double *points, int pointsPerFile, int pointCount, int index){
int newindex= blockIdx.x * blockDim.x + threadIdx.x;
int stride= blockDim.x * gridDim.x;

for(int i=newindex;i<nRxns;i+=stride){
points[pointCount+pointsPerFile*i]=d_prevPoint[nRxns*index+i]+d_stepDist*d_umat[nRxns*index+i];
}
}