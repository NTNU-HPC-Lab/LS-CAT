#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reprojectPoint2(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount,int index){
int newindex= blockIdx.x * blockDim.x + threadIdx.x;
int stride= blockDim.x * gridDim.x;

for(int i=newindex;i<nRxns;i+=stride){
points[pointCount+pointsPerFile*i]=0;
for(int j=0;j<nRxns-istart;j++){
points[pointCount+pointsPerFile*i]+=d_N[j*nRxns+i]*d_umat[nRxns*index+j];//here N*tmp
}
}
}