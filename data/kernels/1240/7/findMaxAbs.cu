#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findMaxAbs(int nRxns, double *d_umat2, int nMets, int *d_rowVec, int *d_colVec, double *d_val, int nnz, double *points, int pointsPerFile, int pointCount, int index){
int newindex = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for(int k=newindex;k<nnz;k+=stride){
d_umat2[nMets*index+d_rowVec[k]]+=d_val[k]*points[pointCount+pointsPerFile*d_colVec[k]];
}

}