#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_1024_one_256(float *A, float *B, float *bnBias, float *bnScale, float *C) {
int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
int ind = line*256 + in_channel;

extern __shared__ float shared_[];
float *weights = shared_ + 1024*4, *output = weights + 256*16, *input = shared_;
float *bias = output + 4*256, *scale = bias + 256;

for (int i = 0; i < 4; i++)
input[ind + i*1024] = A[tile*4096 + i*1024 + ind];
bias[in_channel] = bnBias[in_channel];
scale[in_channel] = bnScale[in_channel];
output[ind] = 0.0f;
__syncthreads();

for (int k = 0; k < 1024; k += 16) {
float *B_start = B + k*256;
for (int i = 0; i < 4; i++)
weights[ind + i*1024] = B_start[i*1024 + ind];
__syncthreads();

float *A_start = input + k;
for (int p = 0; p < 16; p++) {
output[ind] += A_start[line*1024 + p] * weights[in_channel + p*256];
}
__syncthreads();
}

float *C_start = C + tile*1024, res = scale[in_channel] * output[ind] + bias[in_channel];
C_start[ind] = res > 0 ? res : 0;
}