#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_256_OuterProduct_256(float *A, float *B, float *C) {
int Tile = blockIdx.x, Part = blockIdx.y, tX = threadIdx.x, tY = threadIdx.y;
int c_input = tY*256 + tX, c_kernel = c_input, T_offset = (Tile<<12) + (Part<<11) + c_input, B_offset = (Tile<<16) + c_kernel;

extern __shared__ float input[];
float *kernel = input + 2048, *out = kernel + 8192;
int B_stride[32] = {0, 256, 512, 768, 1024, 1280, 1536, 1792, 2048, 2304, 2560, 2816, 3072, 3328, 3584, 3840, 4096, 4352, 4608, 4864, 5120, 5376, 5632, 5888, 6144, 6400, 6656, 6912, 7168, 7424, 7680, 7936};
out[c_input] = 0.0f;
out[c_input+1024] = 0;

input[c_input] = A[T_offset];
input[c_input+1024] = A[T_offset+1024];

for (int k = 0; k < 8; k++) {
int B_start = B_offset + (k<<13); // 32*64
kernel[c_kernel] = B[B_start], kernel[c_kernel+1024] = B[B_start+1024];
kernel[c_kernel+2048] = B[B_start+2048], kernel[c_kernel+3072] = B[B_start+3072];
kernel[c_kernel+4096] = B[B_start+4096], kernel[c_kernel+5120] = B[B_start+5120];
kernel[c_kernel+6144] = B[B_start+6144], kernel[c_kernel+7168] = B[B_start+7168];

__syncthreads();

float sum = 0, sum1 = 0;
int y_tmp = (tY<<8)+(k<<5), y_tmp1 = y_tmp+1024;
for (int j = 0; j < 32; j++) {
sum += input[y_tmp + j] * kernel[tX + B_stride[j]];
sum1 += input[y_tmp1 + j] * kernel[tX + B_stride[j]];
}
out[c_input] += sum;
out[c_input+1024] += sum1;
__syncthreads();
}

C[T_offset] = out[c_input];
C[T_offset+1024] = out[c_input+1024];
}