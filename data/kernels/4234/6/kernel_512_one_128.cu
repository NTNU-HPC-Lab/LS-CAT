#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_512_one_128(float *A, float *B, float *bnBias, float *bnScale, float *C) {
int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
int ind = line*128 + in_channel;

extern __shared__ float shared_[];
float *weights = shared_ + 512*4, *output = weights + 128*64, *input = shared_;
float *bias = output + 4*128, *scale = bias + 128;

for (int i = 0; i < 4; i++)
input[ind + i*512] = A[tile*2048 + i*512 + ind];
bias[in_channel] = bnBias[in_channel];
scale[in_channel] = bnScale[in_channel];
output[ind] = 0.0f;
__syncthreads();

for (int k = 0; k < 512; k += 64) {
float *B_start = B + k*128;
for (int i = 0; i < 16; i++)
weights[ind + i*512] = B_start[i*512 + ind];
__syncthreads();

float *A_start = input + k;
for (int p = 0; p < 64; p++) {
output[ind] += A_start[line*512 + p] * weights[in_channel + p*128];
}
__syncthreads();
}

float *C_start = C + tile*512, res = scale[in_channel] * output[ind] + bias[in_channel];
C_start[ind] = res > 0 ? res : 0;
}