#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_256_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) {
int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
int c_input = Inx*6 + Iny;

__shared__ float bias, scale;
extern __shared__ float input[];

input[c_input] = pInputs[c_input*16*256 + (Tilex*4+Tiley)*256 + kz];
bias = pBiases[kz];
scale = pScales[kz];
__syncthreads();

float tmp = 0;
switch(Inx) {
case 0:
tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
break;
case 1:
tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
break;
case 2:
tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
break;
case 3:
tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
break;
}
__syncthreads();

input[c_input] = tmp;
__syncthreads();

if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;

int x;
float o;
switch(Iny) {
case 0:
x = Inx*6;
o = scale*(input[x]+input[x+1]+input[x+2]+input[x+3]+input[x+4]) + bias;
pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+1)*256 + kz] = o > 0 ? o : 0;
break;
case 1:
x = Inx*6;
o = scale*(input[x+1] - input[x+2] + 2*input[x+3] - 2*input[x+4]) + bias;
pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+2)*256 + kz] = o > 0 ? o : 0;
break;
case 2:
if (Tiley == 3) break;
x = Inx*6;
o = scale*(input[x+1] + input[x+2] + 4*input[x+3] + 4*input[x+4]) + bias;
pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+3)*256 + kz] = o > 0 ? o : 0;
break;
case 3:
if (Tiley == 3) break;
x = Inx*6;
o = scale*(input[x+1] - input[x+2] + 8*input[x+3] - 8*input[x+4] + input[x+5]) + bias;
pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+4)*256 + kz] = o > 0 ? o : 0;
break;
}
}