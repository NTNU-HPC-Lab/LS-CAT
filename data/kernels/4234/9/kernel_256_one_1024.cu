#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_256_one_1024(float *A, float *B, float *bnBias, float *bnScale, float *C) {
int tile = blockIdx.x, part = blockIdx.y, in_channel = threadIdx.x, line = threadIdx.y;
int ind = line*256 + in_channel;

extern __shared__ float shared_[];
float *weights = shared_ + 256*4, *output = weights + 256*32, *input = shared_;
float *bias = output + 4*256, *scale = bias + 256;

input[ind] = A[tile * 1024 + ind];
bias[in_channel] = bnBias[part*256 + in_channel];
scale[in_channel] = bnScale[part*256+ in_channel];
output[ind] = 0.0f;
__syncthreads();

for (int k = 0; k < 256; k += 32) {
for (int i = 0; i < 8; i++)
weights[ind + 1024*i] = B[(k + i*4 + line)*1024 + part*256 + in_channel];
__syncthreads();

float *A_start = input + k;
for (int p = 0; p < 32; p++) {
output[ind] += A_start[line*256 + p] * weights[in_channel + p*256];
}
__syncthreads();
}

float *C_start = C + tile*4096 + part*256;
C_start[line * 1024 + in_channel] = scale[in_channel] * output[ind] + bias[in_channel];
}