#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_128_one_512(float *A, float *B, float *bnBias, float *bnScale, float *C) {
int tile = blockIdx.x, part = blockIdx.y, in_channel = threadIdx.x, line = threadIdx.y;
int ind = line*128 + in_channel;

extern __shared__ float shared_[];
float *weights = shared_ + 128*4, *output = weights + 128*64, *input = shared_;
float *bias = output + 4*128, *scale = bias + 128;

input[ind] = A[tile * 512 + ind];
bias[in_channel] = bnBias[part*128 + in_channel];
scale[in_channel] = bnScale[part*128+ in_channel];
output[ind] = 0.0f;
__syncthreads();

for (int k = 0; k < 128; k += 64) {
for (int i = 0; i < 16; i++)
weights[ind + 512*i] = B[(k + i*4 + line)*512 + part*128 + in_channel];
__syncthreads();

float *A_start = input + k;
for (int p = 0; p < 64; p++) {
output[ind] += A_start[line*128 + p] * weights[in_channel + p*128];
}
__syncthreads();
}

float *C_start = C + tile*2048 + part*128;
float res = scale[in_channel] * output[ind] + bias[in_channel];
C_start[line * 512 + in_channel] = res;
}