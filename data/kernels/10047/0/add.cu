#include "hip/hip_runtime.h"
#include "includes.h"
//nvcc -ptx vectoradd_float.cu -o vectoradd_float.ptx

extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] + b[i];
}
}