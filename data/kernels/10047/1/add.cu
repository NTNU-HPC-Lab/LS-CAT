#include "hip/hip_runtime.h"
#include "includes.h"
//nvcc -ptx vectoradd_double.cu -o vectoradd_double.ptx

extern "C"
__global__ void add(int n, double *a, double *b, double *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] + b[i];
}
}