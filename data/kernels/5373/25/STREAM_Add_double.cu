#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Add_double(double *a, double *b, double *c,  size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
c[idx] = a[idx]+b[idx];
idx   += blockDim.x * gridDim.x;
}
}