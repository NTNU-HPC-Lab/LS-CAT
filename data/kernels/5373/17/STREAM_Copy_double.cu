#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Copy_double(double *a, double *b, size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
b[idx] = a[idx];
idx   += blockDim.x * gridDim.x;
}
}