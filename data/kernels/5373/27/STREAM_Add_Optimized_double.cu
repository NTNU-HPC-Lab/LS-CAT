#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Add_Optimized_double(double *a, double *b, double *c,  size_t len)
{
/*
* Ensure size of thread index space is as large as or greater than
* vector index space else return.
*/
if (blockDim.x * gridDim.x < len) return;
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < len) c[idx] = a[idx]+b[idx];
}