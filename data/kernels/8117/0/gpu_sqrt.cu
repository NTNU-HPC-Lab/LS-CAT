#include "hip/hip_runtime.h"
#include "includes.h"

long N = 6400000000;
int doPrint = 0;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
__global__ void gpu_sqrt(float* a, long N) {
long element = blockIdx.x*blockDim.x + threadIdx.x;
if (element < N) a[element] = sqrt(a[element]);
}