#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repeat_x_for_clusters(float * x,int size)
{
int index = blockIdx.x * blockDim.x + threadIdx.x ;
int thread_index = threadIdx.x ;
int block_index = blockIdx.x ;

if (block_index > 0 && index < size)
{
x[index] = x[thread_index] ;
}
}