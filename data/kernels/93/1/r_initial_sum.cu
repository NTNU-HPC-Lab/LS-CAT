#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);





__global__ void r_initial_sum(float* a , int * indeces , float* x,float * r ,float * r_squared ,float * p_sum ,int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x ;
int local_index = threadIdx.x ;

__shared__ float shared_r_squared[1024] ;
__shared__ float shared_p_sum[1024] ;

shared_r_squared[local_index] = 0 ;
shared_p_sum[local_index] = 0;
__syncthreads() ;

if (index < size)
{
for (int i = 0 ; i<3 ; i++)
{
shared_p_sum[local_index] += a[3*index  + i] * r[indeces[3*index + i]] ;
}
__syncthreads() ;

shared_r_squared[local_index] = r[index] * r[index] ;
shared_p_sum[local_index] = shared_p_sum[local_index] * r[index] ;
}

__syncthreads() ;
for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
{
if (threadIdx.x < s)
{
shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
__syncthreads() ;
}

}

if (threadIdx.x == 0)
{
r_squared[blockIdx.x] = shared_r_squared[0];
p_sum[blockIdx.x] = shared_p_sum[0];
__syncthreads() ;
}
}