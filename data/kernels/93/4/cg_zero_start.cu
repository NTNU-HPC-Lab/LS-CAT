#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cg_zero_start(float* a , float* x,float * b ,int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x ;
int local_index = threadIdx.x ;
int block_index = blockIdx.x ;

__shared__ float shared_r_squared[1024] ;
__shared__ float shared_p_sum[1024] ;
float local_b ;

shared_r_squared[local_index] = 0 ;
shared_p_sum[local_index] = 0;
__syncthreads() ;

if (index < size)
{
local_b = b[index + 2 * block_index + 1] ;

for (int i = 0 ; i<3 ; i++)
{
shared_p_sum[local_index] += a[3*index  + i] * b[index + 2 * block_index +  i] ;
}
__syncthreads() ;

shared_r_squared[local_index] = local_b * local_b ;
shared_p_sum[local_index] = shared_p_sum[local_index] * local_b ;

__syncthreads() ;

for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
{
if (threadIdx.x < s)
{
shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
__syncthreads() ;
}
}
__syncthreads();

float alpha = shared_r_squared[0]/shared_p_sum[0] ;
x[index] = x[index] + alpha * local_b ;
}
}