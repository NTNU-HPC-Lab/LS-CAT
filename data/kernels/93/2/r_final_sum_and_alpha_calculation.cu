#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);





__global__ void r_final_sum_and_alpha_calculation(float * r_squared ,float * p_sum ,int size)
{
int index = threadIdx.x ;

__shared__ float shared_r_squared[1024] ;
__shared__ float shared_p_sum[1024] ;

if (index < size)
{
shared_r_squared[index] = r_squared[index]  ;
shared_p_sum[index] = p_sum[index]  ;
} else
{
shared_r_squared[index] = 0 ;
shared_p_sum[index] = 0 ;
}

__syncthreads() ;

for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
{
if (index < s)
{
shared_r_squared[index] = shared_r_squared[index] + shared_r_squared[index +s] ;
shared_p_sum[index] = shared_p_sum[index] + shared_p_sum[index +s] ;
__syncthreads() ;
}
}
if(threadIdx.x == 0)
{
//alpha
r_squared[blockIdx.x] = shared_r_squared[0]/shared_p_sum[0] ;

}
}