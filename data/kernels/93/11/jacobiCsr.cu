#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobiCsr(float *x, const float *diagonal_values , const float * values, const int *rowPtr ,const int *colIdx,const float *y, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size)
{
int row_start = rowPtr[index];
int row_end = rowPtr[index +1];

float sum = 0 ;

for (int j = 0 ; j< 30 ; j++)
{
for (int i = row_start ; i< row_end ; i++)
{
sum += values[i] * x[colIdx[i]];
}
x[index] =( y[index] - sum )/diagonal_values[index] ;
sum = 0 ;
__syncthreads();
}
}
}