#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x33(float* x34, float* x35, float* x36, int x37) {
int x38 = gridDim.x * blockDim.x;
int x39 = threadIdx.x + blockIdx.x * blockDim.x;
while (x39 < x37) {
int x40 = x39;
x36[x40] = x34[x40] / x35[x40];
x39 = x39 + x38;
}
}