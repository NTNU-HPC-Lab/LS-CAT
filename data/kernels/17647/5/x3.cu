#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x3(int* x4, int x5, int x6) {
int x7 = gridDim.x * blockDim.x;
int x8 = threadIdx.x + blockIdx.x * blockDim.x;
int x9 = -x5;
while (x8 < x6) {
int x10 = x8;
if (x4[x10] > x5) x4[x10] = x5;
if (x4[x10] < x9) x4[x10] = x9;
x8 = x8 + x7;
}
}