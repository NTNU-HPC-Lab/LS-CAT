#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x15(float* x16, float* x17, float* x18, int x19) {
int x20 = gridDim.x * blockDim.x;
int x21 = threadIdx.x + blockIdx.x * blockDim.x;
while (x21 < x19) {
int x22 = x21;
x18[x22] = x16[x22] - x17[x22];
x21 = x21 + x20;
}
}