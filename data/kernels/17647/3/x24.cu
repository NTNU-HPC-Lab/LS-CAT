#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x24(float* x25, float* x26, float* x27, int x28) {
int x29 = gridDim.x * blockDim.x;
int x30 = threadIdx.x + blockIdx.x * blockDim.x;
while (x30 < x28) {
int x31 = x30;
x27[x31] = x25[x31] * x26[x31];
x30 = x30 + x29;
}
}