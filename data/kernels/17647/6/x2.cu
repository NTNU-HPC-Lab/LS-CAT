#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x2(float* x3, float x4, int x5) {
int x6 = gridDim.x * blockDim.x;
int x7 = threadIdx.x + blockIdx.x * blockDim.x;
while (x7 < x5) {
x3[x7] = x4;
x7 = x7 + x6;
}
}