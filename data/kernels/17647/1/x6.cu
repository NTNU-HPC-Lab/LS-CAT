#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x6(float* x7, float* x8, float* x9, int x10) {
int x11 = gridDim.x * blockDim.x;
int x12 = threadIdx.x + blockIdx.x * blockDim.x;
while (x12 < x10) {
int x13 = x12;
x9[x13] = x7[x13] + x8[x13];
x12 = x12 + x11;
}
}