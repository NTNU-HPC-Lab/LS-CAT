#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FloatDivByConstant(float *A, float constant)
{
unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
A[i]=A[i]/constant;
}