#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FloatMul(float *A, float *B, float *C)
{
unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
C[i] = A[i] * B[i];
}