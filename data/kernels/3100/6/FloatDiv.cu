#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FloatDiv(float *A, float *B, float *C)
{
unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;

if (B[i] != 0) {
C[i] = A[i] / B[i];
}
else {
C[i] = 0;
}

}