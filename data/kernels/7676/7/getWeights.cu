#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getWeights( const double* restoredEigenvectors , const double* meanSubtractedImages , double* weights , std::size_t imageNum , std::size_t pixelNum , std::size_t componentNum )
{
std::size_t row = blockIdx.x;
std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

if(col >= imageNum || row >= componentNum)
{
return;
}

weights[row * imageNum + col] = 0.0;
for(std::size_t i = 0; i < pixelNum; ++i)
{
weights[row * imageNum + col] += restoredEigenvectors[row * pixelNum + i] * meanSubtractedImages[col * pixelNum + i];
}
}