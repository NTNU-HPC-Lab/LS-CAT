#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getDistances( const double* trainingWeights , const double* testWeights , double* distances , std::size_t trainImageNum , std::size_t testImageNum , std::size_t componentNum )
{
std::size_t row = blockIdx.x;
std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

if(col >= testImageNum || row >= trainImageNum)
{
return;
}

distances[row * testImageNum + col] = 0.0;
for(std::size_t i = 0; i < componentNum; ++i)
{
distances[row * testImageNum + col] += fabs(trainingWeights[i * trainImageNum + row] - testWeights[i * testImageNum + col]);
}
}