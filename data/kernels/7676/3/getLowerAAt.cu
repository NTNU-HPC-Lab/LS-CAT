#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getLowerAAt( const double* A, double* S, std::size_t imageNum, std::size_t pixelNum )
{
std::size_t row = blockIdx.x;
std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

if(row >= imageNum || col >= imageNum)
{
return;
}

S[row * imageNum + col] = 0.0;
for(std::size_t i = 0; i < pixelNum; ++i)
{
S[row * imageNum + col] += A[row * pixelNum + i] * A[col * pixelNum + i];
}
}