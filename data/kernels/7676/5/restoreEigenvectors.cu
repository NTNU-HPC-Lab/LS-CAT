#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void restoreEigenvectors( const double* meanSubtractedImages , const double* reducedEigenvectors , double* restoredEigenvectors , std::size_t imageNum , std::size_t pixelNum , std::size_t componentNum )
{
std::size_t row = blockIdx.x;
std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

if(col >= pixelNum || row >= componentNum)
{
return;
}

restoredEigenvectors[row * pixelNum + col] = 0.0;
for(std::size_t i = 0; i < imageNum; ++i)
{
restoredEigenvectors[row * pixelNum + col] += reducedEigenvectors[(imageNum - row - 1) * imageNum + i] * meanSubtractedImages[i * pixelNum + col];
}
}