#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getTestWeights( const double* restoredEigenvectors , const double* meanImage , const double* testImages , double* testWeights , std::size_t testImageNum , std::size_t pixelNum , std::size_t componentNum )
{
std::size_t row = blockIdx.x;
std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

if(col >= testImageNum || row >= componentNum)
{
return;
}

testWeights[row * testImageNum + col] = 0.0;
for(std::size_t i = 0; i < pixelNum; ++i)
{
double testImagePixelValue = testImages[col * pixelNum + i] - meanImage[i];
if(testImagePixelValue < 0.0)
{
testImagePixelValue = 0.0;
}
testWeights[row * testImageNum + col] += restoredEigenvectors[row * pixelNum + i] * (testImagePixelValue);
}
}