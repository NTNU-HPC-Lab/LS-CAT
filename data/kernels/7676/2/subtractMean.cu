#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void subtractMean( double* images, const double* meanImage, std::size_t imageNum, std::size_t pixelNum )
{
std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;

if(col >= pixelNum)
{
return;
}

for(std::size_t row = 0; row < imageNum; ++row)
{
images[row*pixelNum + col] -= meanImage[col];

if(images[row*pixelNum + col] < 0.0)
{
images[row*pixelNum + col] = 0.0;
}
}
}