#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getMeanImage( const double* images, double* meanImage, std::size_t imageNum, std::size_t pixelNum )
{
std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;

if(col >= pixelNum)
{
return;
}

meanImage[col] = 0.0;
for(std::size_t row = 0; row < imageNum; ++row)
{
meanImage[col] += images[row*pixelNum + col];
}

meanImage[col] /= imageNum;
}