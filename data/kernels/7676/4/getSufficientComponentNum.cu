#include "hip/hip_runtime.h"
#include "includes.h"



// helper for CUDA error handling
__global__ void getSufficientComponentNum(const double* eigenvalues, std::size_t* componentNum, std::size_t eigenRows, double epsilon)
{
double variance = 0;
for(std::size_t i = 0; i < eigenRows; ++i)
{
variance += eigenvalues[i];
}
variance *= eigenRows;

(*componentNum) = 1;
double subVariance = eigenRows * eigenvalues[eigenRows - 1];
double explanatoryScore = subVariance / variance;
for(; (*componentNum) < eigenRows && explanatoryScore <= epsilon; (*componentNum) += 1)
{
subVariance += eigenRows * eigenvalues[eigenRows - (*componentNum) - 1];
explanatoryScore = subVariance / variance;
}
}