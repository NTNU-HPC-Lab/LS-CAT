#include "hip/hip_runtime.h"
#include "includes.h"


__device__ double caculateValueOfWeight(double parameter, int sign, double alpha)
{
return (parameter*sign*alpha);
}
__global__ void updateWeights(double* weights, double* parameters,double* otherp, int sign, double alpha)
{
int index = threadIdx.x;
double value = weights[index];
weights[index] = value + caculateValueOfWeight( parameters[index], sign, alpha);

}