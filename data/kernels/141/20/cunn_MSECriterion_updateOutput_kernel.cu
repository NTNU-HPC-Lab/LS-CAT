#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
__shared__ float buffer[MSECRITERION_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *target_k = target + k*dim;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

// mse
buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float z = input_k[i] - target_k[i];
buffer[threadIdx.x] += z*z;
}
__syncthreads();


//reduce
if (threadIdx.x == 0)
{
*output = 0;
for (int i=0; i<blockDim.x; i++)
{
*output += buffer[i];
}
if (sizeAverage)
*output /= dim;
}
}