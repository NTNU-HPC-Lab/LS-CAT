#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
// iterators
int xx, yy;

// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
gradOutput = gradOutput + o*output_w*output_h;
gradInput = gradInput + i*input_w*input_h;

// compute gradInput
for(yy = yy_start; yy < yy_end; yy+=yy_step) {
for(xx = xx_start; xx < xx_end; xx+=xx_step) {
float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
float *ptr_gradOutput = gradOutput + yy*output_w + xx;
float z = *ptr_gradOutput;
int kx, ky;
for(ky = 0; ky < kH; ky++) {
for(kx = 0; kx < kW; kx++)
ptr_gradInput[kx] += z / float(kW*kH);
ptr_gradInput += input_w;
}
}
}
}