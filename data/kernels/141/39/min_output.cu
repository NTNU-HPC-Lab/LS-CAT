#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
// output offset:
long o = threadIdx.x + blockDim.x * blockIdx.x;
if (o >= nrows) return;

// input offset:
long i = o * ncols;

// move pointers
input = input + i;

// compute min:
float min = input[0];
long argmin = 0;
long ii;
for (ii=1; ii<ncols; ii++) {
float val = input[ii];
if (val < min) {
min = val;
argmin = ii;
}
}

// store
output[o] = min;
indices[o] = argmin+1;
}