#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
int i = threadIdx.x;
C[i] = A[i] + B[i];
}