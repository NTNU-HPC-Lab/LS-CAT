#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045





__global__ void Inicializacao( double *uprev, const int n ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
double x = idx * dx;
if( idx < n + 1 ) {
if( x <= 0.5 ) {
uprev[ idx ] = 200 * x;
}
else {
uprev[ idx ] = 200 * ( 1. - x );
}
}
}