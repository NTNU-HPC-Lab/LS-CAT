#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045





__global__ void Atualiza( double *u, double *u_prev, const int n ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if( idx == 0 ) {
u[ 0 ] = u[ n ] = 0.; /* forca condicao de contorno */
}
else if( idx < n ) {
u[ idx ] = u_prev[ idx ] + kappa * dt / ( dx * dx ) * ( u_prev[ idx - 1 ] - 2 * u_prev[ idx ] + u_prev[ idx + 1 ] );
}
}