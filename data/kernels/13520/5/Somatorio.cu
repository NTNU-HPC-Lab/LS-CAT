#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Somatorio( float *input, float *results, long int n ) {
extern __shared__ float sdata[];
int idx = blockIdx.x * blockDim.x + threadIdx.x, tx = threadIdx.x;
float x = 0.;
if( idx < n ) {
x = input[ idx ];
}
sdata[ tx ] = x;
__syncthreads( );
for( int offset = blockDim.x / 2; offset > 0; offset >>= 1 ) {
if( tx < offset ) {
sdata[ tx ] += sdata[ tx + offset ];
}
__syncthreads( );
}
if( threadIdx.x == 0 ) {
results[ blockIdx.x ] = sdata[ 0 ];
}
}