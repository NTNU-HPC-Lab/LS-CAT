#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Evolve( int *val, int *aux, int n ) {
int up, upright, right, rightdown, down, downleft, left, leftup;
int sum = 0, estado;
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if ( i > 0 && i < (n - 1) && j > 0 && j < (n - 1) ){
estado = val[ i * n + j ];
up = val[ ( i - 1 ) * n + j ];
upright = val[ ( i - 1 ) * n + j + 1 ];
right = val[ i * n + j + 1 ];
rightdown = val[ ( i + 1 ) * n + j + 1 ];
down = val[ ( i + 1 ) * n + j ];
downleft = val[ ( i + 1 ) * n + j - 1 ];
left = val[ i * n + j - 1 ];
leftup = val[ ( i - 1 ) * n + j - 1 ];
sum = up + upright + right + rightdown + down + downleft + left + leftup;
if( sum == 3 ) {
estado = 1;
}
else if( ( estado == 1 ) && ( ( sum < 2 ) || ( sum > 3 ) ) ) {
estado = 0;
}
aux[ i * n + j ] = estado;
}
}