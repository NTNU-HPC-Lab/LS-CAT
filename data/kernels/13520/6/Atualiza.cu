#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Atualiza( double *u, double *u_prev, const int n ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if( idx == 0 ) {
u[ 0 ] = u[ n ] = 0.; /* forca condicao de contorno */
}
else if( idx < n ) {
u[ idx ] = u_prev[ idx ] + kappa * dt / ( dx * dx ) * ( u_prev[ idx - 1 ] - 2 * u_prev[ idx ] + u_prev[ idx + 1 ] );
}
}