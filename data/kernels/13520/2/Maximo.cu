#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045





__global__ void Maximo( double *input, double *results, int n ) {
extern __shared__ double sdata[];
int idx = blockIdx.x * blockDim.x + threadIdx.x, tx = threadIdx.x;
double x = 0.;
if( idx < n ) {
x = input[ idx ];
}
sdata[ tx ] = x;
__syncthreads( );
for( int offset = blockDim.x / 2; offset > 0; offset >>= 1 ) {
if( tx < offset ) {
if( sdata[ tx ] < sdata[ tx + offset ] ) {
sdata[ tx ] = sdata[ tx + offset ];
}
}
__syncthreads( );
}
if( threadIdx.x == 0 ) {
results[ blockIdx.x ] = sdata[ 0 ];
}
}