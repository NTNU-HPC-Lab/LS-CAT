#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixAdd_d( float *A, float *B, float *C, int N ) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
int index = i * N + j;
if( ( i < N ) && ( j < N ) ) {
C[ index ] = A[ index ] + B[ index ];
}
}