#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Evolve( int *val, int *aux, int n ) {
int up, upright, right, rightdown, down, downleft, left, leftup;
int sum = 0, estado;
const int tx = threadIdx.x + 1, ty = threadIdx.y + 1;
const int i = blockIdx.y * blockDim.y + threadIdx.y;
const int j = blockIdx.x * blockDim.x + threadIdx.x;
const int b2 = BSIZE + 2;
__shared__ float sdata[ b2 ][ b2 ];

sdata[ ty ][ tx ] = val[ i * n + j ];
if( ( tx == 1 ) && ( ty == 1 ) ) {
int stx = blockIdx.x * blockDim.x;
int sty = blockIdx.y * blockDim.y;
sdata[ 0      ][ 0      ] = val[ ( sty - 1     ) * n + stx - 1     ];
sdata[ 0      ][ b2 - 1 ] = val[ ( sty - 1     ) * n + stx + BSIZE ];
sdata[ b2 - 1 ][ 0      ] = val[ ( sty + BSIZE ) * n + stx - 1     ];
sdata[ b2 - 1 ][ b2 - 1 ] = val[ ( sty + BSIZE ) * n + stx + BSIZE ];
}
if( ( j > 0 ) && ( tx == 1 ) ) {
sdata[ ty     ][ 0      ] = val[ i * n + j - 1 ];
}
if( ( j < ( n - 1 ) ) && ( tx == BSIZE ) ) {
sdata[ ty     ][ b2 - 1 ] = val[ i * n + j + 1 ];
}
if( ( i > 0 ) && ( ty == 1 ) ) {
sdata[ 0      ][ tx     ] = val[ ( i - 1 ) * n + j ];
}
if( ( i < ( n - 1 ) ) && ( ty == BSIZE ) ) {
sdata[ b2 - 1 ][ tx     ] = val[ ( i + 1 ) * n + j ];
}
__syncthreads( );
if( ( i > 0 ) && ( i < ( n - 1 ) ) && ( j > 0 ) && ( j < ( n - 1 ) ) ) {
estado = sdata[ ty ][ tx ];
up = sdata[ ty - 1 ][ tx ];
upright = sdata[ ty - 1 ][ tx + 1 ];
right = sdata[ ty ][ tx + 1 ];
rightdown = sdata[ ty + 1 ][ tx + 1 ];
down = sdata[ ty + 1 ][ tx ];
downleft = sdata[ ty + 1 ][ tx - 1 ];
left = sdata[ ty ][ tx - 1 ];
leftup = sdata[ ty - 1 ][ tx - 1 ];
sum = up + upright + right + rightdown + down + downleft + left + leftup;
if( sum == 3 ) {
estado = 1;
}
else if( ( estado == 1 ) && ( ( sum < 2 ) || ( sum > 3 ) ) ) {
estado = 0;
}
aux[ i * n + j ] = estado;
}
}