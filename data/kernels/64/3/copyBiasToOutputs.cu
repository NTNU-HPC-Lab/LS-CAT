#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyBiasToOutputs(float *ptrbias, float *ptroutput, const int size1, const int size2, const int nOutputPlane, const int linestride, const int imstride)
{
// each thread has a value to manage...
//const int blk =blockDim.x;
const int tidx=blockDim.x*blockIdx.x + threadIdx.x;
const int tidy=blockIdx.y;
const int tidz=blockIdx.z;

float val = ptrbias[tidx];
ptroutput+= tidz*imstride + tidy*linestride;

for(int k=0; k<size2; k++)
{
if(tidx<nOutputPlane) {
ptroutput[k*nOutputPlane+tidx]=val;
}
}
}