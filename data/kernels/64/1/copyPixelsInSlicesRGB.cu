#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyPixelsInSlicesRGB(float *ptrinput0, float *ptrkslices0, int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown, int inputstr0, int kslicesstr0, int batchsize)
{
// each block does one pixel of the input image
// each kernel slice is represented by its upper-left coordinates

const int pixi=blockIdx.x;
const int pixj=blockIdx.y*blockDim.y + threadIdx.y;
const int tidx=threadIdx.x;
const int batchindex=blockIdx.z*blockDim.z+threadIdx.z;

int i,j;

int imin, jmin, imax, jmax;
int inputoffset, ksliceoffset;

// step 1 : find which kernel slices contain the values of the pixel
__shared__ int _imin, _jmin[32], _imax, _jmax[32], _inputoffset[32][3], _ksliceoffset[32][3];
if(threadIdx.z==0)
{
imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;
if(threadIdx.x==0 && threadIdx.y==0)
{
_imin=imin;
_imax=imax;
}
if(threadIdx.x==0)
{
_jmin[threadIdx.y]=jmin;
_jmax[threadIdx.y]=jmax;
}
inputoffset = inputstr0*blockIdx.z*blockDim.z + ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
ksliceoffset= kslicesstr0*blockIdx.z*blockDim.z + ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;
_inputoffset[threadIdx.y][threadIdx.x]=inputoffset;
_ksliceoffset[threadIdx.y][threadIdx.x]=ksliceoffset;
}

__syncthreads();

if(batchindex >= batchsize) return;
if(pixj > isize2 + padleft + padright -1) return;


if(threadIdx.z>0)
{
imin=_imin;
imax=_imax;
jmin=_jmin[threadIdx.y];
jmax=_jmax[threadIdx.y];
inputoffset=_inputoffset[threadIdx.y][threadIdx.x];
ksliceoffset=_ksliceoffset[threadIdx.y][threadIdx.x];
}

// step 2 : move the pointers
// this one goes to where the pixel is at
ptrinput0   += inputoffset+inputstr0*threadIdx.z ;
ptrkslices0 += ksliceoffset+kslicesstr0*threadIdx.z ;

const int stridej = (kH*kW - dW) * nInputPlane;
const int stridei = (size2*kH-dH) * kW *nInputPlane - (jmax-jmin+1) * stridej ;

bool zeropad = pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;


// read pixel
// load the stuff first...
//for (b=0; b<batchsize; b++)
//{
float * ptrinput    = ptrinput0;
float * ptrkslices  = ptrkslices0;

float pixvalue;
if (zeropad) 	{
pixvalue=0;
}
else	{
pixvalue=ptrinput[tidx];
}


//	write to memory
for(i=imin; i<imax+1; i++) {
for(j=jmin; j<jmax+1; j++) {
if(zeropad)
{
ptrkslices[tidx]=0;
}
else {
ptrkslices[tidx]=pixvalue;
}
ptrkslices += stridej;
}
ptrkslices += stridei;
}
//}
}