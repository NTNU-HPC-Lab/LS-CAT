#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "copyPixelsInSlicesRGB.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *ptrinput0 = NULL;
hipMalloc(&ptrinput0, XSIZE*YSIZE);
float *ptrkslices0 = NULL;
hipMalloc(&ptrkslices0, XSIZE*YSIZE);
int dH = 1;
int dW = 1;
int kH = 1;
int kW = 1;
int size1 = XSIZE*YSIZE;
int size2 = XSIZE*YSIZE;
int isize1 = XSIZE*YSIZE;
int isize2 = XSIZE*YSIZE;
int nInputPlane = 1;
int padleft = 1;
int padright = 1;
int padup = 1;
int paddown = 1;
int inputstr0 = 1;
int kslicesstr0 = 1;
int batchsize = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
copyPixelsInSlicesRGB<<<gridBlock,threadBlock>>>(ptrinput0,ptrkslices0,dH,dW,kH,kW,size1,size2,isize1,isize2,nInputPlane,padleft,padright,padup,paddown,inputstr0,kslicesstr0,batchsize);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
copyPixelsInSlicesRGB<<<gridBlock,threadBlock>>>(ptrinput0,ptrkslices0,dH,dW,kH,kW,size1,size2,isize1,isize2,nInputPlane,padleft,padright,padup,paddown,inputstr0,kslicesstr0,batchsize);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
copyPixelsInSlicesRGB<<<gridBlock,threadBlock>>>(ptrinput0,ptrkslices0,dH,dW,kH,kW,size1,size2,isize1,isize2,nInputPlane,padleft,padright,padup,paddown,inputstr0,kslicesstr0,batchsize);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}