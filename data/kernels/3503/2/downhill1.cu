#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright [2019] [illava(illava@outlook.com)]

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/




const int blockSize = 1024;

// The original code

// d_key is the original key, keeps untouched

// d_temp is a copy of d_key, changes during algorithm

// shift = 2 ^ d

// __global__ void uphill(uint32_t *d_value, uint8_t *d_key, uint8_t *d_temp,
//                        int64_t n, int64_t shift)
// {
//     int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
__global__ void downhill1(uint32_t *d_value, uint8_t *d_key, uint8_t *d_temp, int64_t n, int64_t shift)
{
int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx + shift < n)
{
if (idx % (2 * shift) == 0)
{
uint32_t tmp                   = d_value[n - 1 - (idx + shift)];
d_value[n - 1 - (idx + shift)] = d_value[n - 1 - idx];
if (d_key[n - 1 - (idx + shift - 1)] == 1)
d_value[n - 1 - idx] = 0;
else if (d_temp[n - 1 - (idx + shift)] == 1)
d_value[n - 1 - idx] = tmp;
else
d_value[n - 1 - idx] = tmp + d_value[n - 1 - idx];
d_temp[n - 1 - (idx + shift)] = 0;
}
}
}