#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright [2019] [illava(illava@outlook.com)]

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/




const int blockSize = 1024;

// The original code

// d_key is the original key, keeps untouched

// d_temp is a copy of d_key, changes during algorithm

// shift = 2 ^ d

// __global__ void uphill(uint32_t *d_value, uint8_t *d_key, uint8_t *d_temp,
//                        int64_t n, int64_t shift)
// {
//     int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
__global__ void assign(uint32_t *x, uint32_t n) { x[0] = n; }