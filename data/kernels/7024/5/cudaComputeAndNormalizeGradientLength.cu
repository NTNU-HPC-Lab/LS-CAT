#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient, int chunk_size_per_thread) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
for (int i = index * chunk_size_per_thread; i < (index + 1) * chunk_size_per_thread; i++) {
int gradient_length = int(sqrt(float(x_gradient[i] * x_gradient[i] + y_gradient[i] * y_gradient[i])));
if (gradient_length > 255) {
gradient_length = 255;
}
channel_values[i] = gradient_length;
}
return;
}