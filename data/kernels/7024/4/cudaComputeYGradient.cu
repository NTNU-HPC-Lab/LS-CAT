#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
int y_kernel[3][3] = { { 1, 2, 1 }, { 0, 0, 0 }, { -1, -2, -1 } };
int index = blockIdx.x * blockDim.x + threadIdx.x;
for (int i = index * chunk_size_per_thread; i < (index + 1) * chunk_size_per_thread - 1; i++) {
if (i + 2 * image_width + 1 < image_width * image_height) {
if (i == 0 && blockIdx.x == 0 && blockIdx.x == 0) {
continue;
} else {
y_gradient[i] =
y_kernel[0][0] * channel[i - 1] +
y_kernel[1][0] * channel[i] +
y_kernel[2][0] * channel[i + 1] +
y_kernel[0][1] * channel[i + image_width - 1] +
y_kernel[1][1] * channel[i + image_width] +
y_kernel[2][1] * channel[i + image_width + 1] +
y_kernel[0][2] * channel[i + 2 * image_width - 1] +
y_kernel[1][2] * channel[i + 2 * image_width] +
y_kernel[2][2] * channel[i + 2 * image_width + 1];
}
}
}
return;
}