#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int xy = i % size;
int b = i / size;

if (b < batches) {
for (int c = 0; c < channels; ++c) {
output[xy + size*(c + channels*b)] += alpha * gt_gpu[i] * a_avg_gpu[i];
//output[xy + size*(c + channels*b)] += gt_gpu[i] * a_avg_gpu[i];
//output[xy + size*(c + channels*b)] += gt_gpu[i] * output[xy + size*(c + channels*b)];
//output[xy + size*(c + channels*b)] = a_avg_gpu[i];
}
}
}