#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repack_input_kernel_2(float *input, float *re_packed_input, int w, int h, int c)
{
//__shared__ uint32_t tmp[33 * 32];  // 33x32 is misaligned 32 x 32 to avoid bank conflicts

int index = blockIdx.x*blockDim.x + threadIdx.x;

const int items_per_channel = w * h;

int c_pack = index % 32;
int chan_index = index / 32;
int chan = (chan_index * 32) % c;
int i = (chan_index * 32) / c;

//for (chan = 0; chan < c; chan += 32)
{
//for (i = 0; i < items_per_channel; ++i)
if (i < items_per_channel)
{
//for (c_pack = 0; c_pack < 32; ++c_pack)
{
float src = input[(chan + c_pack)*items_per_channel + i];

re_packed_input[chan*items_per_channel + i * 32 + c_pack] = src;
}
}
}
}