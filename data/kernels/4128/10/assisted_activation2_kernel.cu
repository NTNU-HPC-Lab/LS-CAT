#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int xy = i % size;
int b = i / size;
float beta = 1 - alpha;

if (b < batches) {
for (int c = 0; c < channels; ++c) {
if(gt_gpu[i] == 0)
output[xy + size*(c + channels*b)] *= beta;

}
}
}