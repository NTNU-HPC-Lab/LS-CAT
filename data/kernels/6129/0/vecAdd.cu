#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c

float* internalBuffer;
int nBuf;

__global__ void vecAdd(float *a, float *b, float *c, int n)
{
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
c[id] = a[id] + b[id];
}