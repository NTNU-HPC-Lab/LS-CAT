#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_CriterionFilter_updateOutput_kernel( float *target, float *ignored_label, int bound, int batch_size, int map_nelem, int blocks_per_sample)
{
int i;
int sample = blockIdx.x / blocks_per_sample;
int step = blockDim.x * blocks_per_sample;
int toffset = sample * map_nelem;
int ignored_label_num = (int)(ignored_label[0]);
for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x; i < map_nelem; i += step) {
if (target[toffset + i] == ignored_label_num) {
target[toffset + i] = (float) bound + 1;
}
}
}