#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_CriterionFilter_updateGradInput_kernel( float *gradInput, float *target, float *ignored_label, int batch_size, int n_classes, int map_nelem, int blocks_per_sample)
{
int i, t;
int sample = blockIdx.x / blocks_per_sample;
int step = blockDim.x * blocks_per_sample;
int toffset = sample * map_nelem;
int ioffset = sample * map_nelem * n_classes;
int ignored_label_num = (int)(ignored_label[0]);
for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x; i < map_nelem; i += step) {
t = (int)target[toffset + i];
if (t == ignored_label_num) {
int j;
for (j = 0; j < n_classes; j++) gradInput[ioffset + j * map_nelem + i] = 0;
}
}
}