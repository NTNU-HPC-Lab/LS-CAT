#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_ld(unsigned *in, unsigned *out){
__shared__ int temp[BLOCK_SIZE + 2*RADIUS];
int gindex = threadIdx.x + blockIdx.x * blockDim.x;
int lindex = threadIdx.x;

temp[lindex] = in[gindex];

if(threadIdx.x < RADIUS){
temp[lindex - RADIUS]     = in[gindex - RADIUS];
temp[lindex + BLOCK_SIZE] = in[gindex - BLOCK_SIZE];
}


__syncthreads();

int result = 0;
for(int offset = -RADIUS; offset < RADIUS; offset++){
result += temp[lindex + offset];
}


out[gindex] = result;

}