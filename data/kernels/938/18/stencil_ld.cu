#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_ld(unsigned *X, unsigned *out, int width, int height){
int kernel[3][3] = { 0, -1, 0,
-1, 5, -1,
0, -1, 0};
int cikti;
int x  = blockIdx.y*32*width + blockIdx.x*32 + threadIdx.y*width + threadIdx.x; //current pixel

//if(x/width<1 || x/width>height-1 || x%width == width-1 || x%width == 1) return; // kenar noktalarinda

cikti  =(kernel[0][0]*X[x-width-1]       +
kernel[0][1]*X[x-width]         +
kernel[0][2]*X[x-width+1]       +
kernel[1][0]*X[x-1]             +
kernel[1][1]*X[x]               +
kernel[1][2]*X[x+1]             +
kernel[2][0]*X[x+width-1]       +
kernel[2][1]*X[x+width-1]       +
kernel[2][2]*X[x+width-1]);


if(cikti < 0)        out[x] = 0;
else if(cikti > 255) out[x] = 255;
else                 out[x] = cikti;

}