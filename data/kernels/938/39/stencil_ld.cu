#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_ld(unsigned *in, unsigned *out){
printf("Thread %d : %d\n", threadIdx.x, in[threadIdx.x]);
out[threadIdx.x] = 2 * in[threadIdx.x];
printf("out location : %p\n", out+threadIdx.x);
printf("in %d : %d\n" , threadIdx.x, in[threadIdx.x]);
printf("out %d : %d\n", threadIdx.x, out[threadIdx.x]);

__syncthreads();

/*
__shared__ int temp[BLOCK_SIZE + 2*RADIUS];
int gindex = threadIdx.x + blockIdx.x * blockDim.x;
int lindex = threadIdx.x + RADIUS;

temp[lindex] = in[gindex];

if(threadIdx.x < RADIUS){
temp[lindex - RADIUS]     = in[gindex - RADIUS];
temp[lindex + BLOCK_SIZE] = in[gindex - BLOCK_SIZE];
}

int result = 0;
for(int offset = -RADIUS; offset < RADIUS; offset++){
result += temp[lindex + offset];
}

__syncthreads();

out[gindex] = result;
*/
}