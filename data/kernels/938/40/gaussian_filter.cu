#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaussian_filter(unsigned *in, unsigned *out, int width, int height){
__shared__ int cikti;
cikti = 0;

__syncthreads();

cikti += in[blockIdx.y*width*2 + blockIdx.x*2 + threadIdx.y*width + threadIdx.x];

__syncthreads();

out[blockIdx.y*width/2 + blockIdx.x] = cikti; // ciktiyi bir sayiya boldugumde garip bir sekilde resim karariyor???(oysa 4 sayiyi topluyoruz, neden ort almiyoruz???)

}