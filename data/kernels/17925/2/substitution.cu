#include "hip/hip_runtime.h"
#include "includes.h"

// Helper function for using CUDA to call kernel functions
hipError_t cuda_code(float* , float*, int , int );
__device__ float sum = 0;

__global__ void substitution(int i, int N, float *row, float *matrix, float*resultVector) {
int j = i + blockIdx.x * blockDim.x + threadIdx.x;
//From previous line, "i" assigns the initial thread index, so threads are not
//created for indexes that will not affect the results
int ij;		//element i,j of the matrix
if (j > i && j < N)
{
ij = j + (N + 1)*i;
row[j] = matrix[ij] * resultVector[j];
atomicAdd(&sum, row[j]);
}
__syncthreads();//Barrier to wait all threads to finish their tasks
}