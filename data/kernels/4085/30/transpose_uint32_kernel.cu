#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
//l.bit_align - algined (n) by 32
//new_ldb - aligned (k) by 256
int index = blockIdx.x*blockDim.x + threadIdx.x;

//for (i = 0; i < src_h; i += 1)
int i = index % src_h;  // l.size*l.size*l.c;
{
//for (j = 0; j < src_w; j += 1)
int j = index / src_h;  // out_h*out_w;
if (j < src_w)
{
((uint32_t *)dst)[j*dst_align / 32 + i] = ((uint32_t *)src)[i*src_align + j];
}
}
}