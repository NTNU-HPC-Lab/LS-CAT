#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= N) return;
int in_index = i;
int in_w = i%w;
i = i / w;
int in_h = i%h;
i = i / h;
int in_c = i%c;
i = i / c;
int b = i%batch;

int out_c = c / (stride*stride);

int c2 = in_c % out_c;
int offset = in_c / out_c;
int w2 = in_w*stride + offset % stride;
int h2 = in_h*stride + offset / stride;

int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

out[in_index] = x[out_index];
}