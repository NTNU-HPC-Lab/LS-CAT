#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (f >= n) return;
int i = 0;
float mean = 0;
for (i = 0; i < size; ++i) {
mean += fabs(weights[f*size + i]);
}
mean = mean / size;
for (i = 0; i < size; ++i) {
binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
//binary[f*size + i] = weights[f*size + i];
}
}