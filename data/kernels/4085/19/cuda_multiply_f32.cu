#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_multiply_f32(float *input_output, size_t size, float multipler)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size) input_output[idx] = input_output[idx] * multipler; // 7-bit (1-bit sign)

}