#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_int8_to_f32(int8_t* input_int8, size_t size, float *output_f32, float multipler)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size) output_f32[idx] = input_int8[idx] * multipler; // 7-bit (1-bit sign)

}