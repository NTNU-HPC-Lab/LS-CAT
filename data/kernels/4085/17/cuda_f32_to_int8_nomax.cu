#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_f32_to_int8_nomax(float* input_f32, size_t size, int8_t *output_int8, float multipler)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size) output_int8[idx] = input_f32[idx] * multipler; // 7-bit (1-bit sign)

}