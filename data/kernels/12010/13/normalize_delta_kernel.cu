#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;
int f = (index/spatial)%filters;

delta[index] = delta[index] * 1./(sqrt(variance[f]) + .000001f) + variance_delta[f] * 2. * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}