#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) {
if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
}
}