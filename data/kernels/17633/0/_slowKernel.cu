#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright (c) Facebook, Inc. and its affiliates.
* All rights reserved.
*
* This source code is licensed under the BSD-style license found in the
* LICENSE file in the root directory of this source tree.
*/



__global__ void _slowKernel(char* ptr, int sz) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
for (; idx < sz; idx += (gridDim.x * blockDim.x)) {
for (int i = 0; i < 100000; ++i) {
ptr[idx] += ptr[(idx + 1007) % sz] + i;
}
}
}