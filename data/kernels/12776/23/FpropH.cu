#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FpropH(float* layer1, const float* synH, const int offset)
{
int i = blockDim.x*blockIdx.x + threadIdx.x; //256
int j = blockDim.y*blockIdx.y + threadIdx.y; //256
atomicAdd(&layer1[256*offset + j], layer1[256*(offset-1) + i] * synH[i*256 + j]);
//__syncthreads();
//if (i == 0)
//   layerH[j] = layer1[j];
}