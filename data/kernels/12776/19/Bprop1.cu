#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Bprop1(const float* in, float* dsyn1, const float* dlayer1, const float alpha)
{
int i = blockDim.y*blockIdx.y + threadIdx.y; //28*28
int j = threadIdx.x;                         //256
int k = blockIdx.x;                          //Data.count

atomicAdd(&dsyn1[i*256 + j], dlayer1[k*256 + j] * in[k*28*28 + i] * alpha);
}