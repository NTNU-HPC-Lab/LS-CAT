#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop2(const float* layer1, const float* syn2, float* out)
{
int i = blockDim.y*blockIdx.y + threadIdx.y; //10
int j = blockIdx.x;  //Data.count
//int k = threadIdx.x; //256
float x = 0.0;
for (int k=0; k < 256; ++k)
x += layer1[j*256 + k] * syn2[k*10 + i];
out[j*10 + i] = x;
}