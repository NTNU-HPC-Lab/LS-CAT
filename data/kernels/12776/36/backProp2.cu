#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backProp2(float* layer1, float* dsyn2, float* label, float* out)
{
int j = blockDim.x*blockIdx.x + threadIdx.x;
int k = blockDim.y*blockIdx.y + threadIdx.y;
float delta = (label[k] - out[k]) * (out[k]*(1.0-out[k]));
dsyn2[j*10 + k] += delta * layer1[j] / (60000.0/10.0);
}