#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop1(const float* in, const float* syn1, float* layer1)
{
int i = threadIdx.x;                         //256
//int j = blockDim.y*blockIdx.y + threadIdx.y; //28*28
int k = blockIdx.x;                          //Data.count
float x = 0.0;
for (int j=0; j < 28*28; ++j)
x += in[k*28*28 + j] * syn1[j*256 + i];
layer1[k*256 + i] = x;
}