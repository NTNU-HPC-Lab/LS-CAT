#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop2(const float* layer1, const float* syn2, float* out, const int offset)
{
int i = blockDim.x*blockIdx.x + threadIdx.x; //4
//int j = blockIdx.x;  //Data.count
int k = blockDim.y*blockIdx.y + threadIdx.y; //256
atomicAdd(&out[i], layer1[256*offset + k] * syn2[k*4 + i]);
}