#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Bprop1(const float* dlayer1, const float* dlayer1i, const float* dlayer1o, const float* in, float* dsyn1, float* dsyn1i, float* dsyn1o, const float alpha)
{
int i = blockDim.y*blockIdx.y + threadIdx.y; //64
int j = threadIdx.x;                         //256
int k = blockIdx.x;                          //Data.count

atomicAdd(&dsyn1[i*256 + j],  dlayer1[k*256 + j]  * in[k*64 + i] * alpha);
atomicAdd(&dsyn1i[i*256 + j], dlayer1i[k*256 + j] * in[k*64 + i] * alpha);
atomicAdd(&dsyn1o[i*256 + j], dlayer1o[k*256 + j] * in[k*64 + i] * alpha);
}