#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Ecalc2(float* out, const float* label)
{
int i = blockDim.x*blockIdx.x + threadIdx.x; //10 * Data.count
out[i] = label[i] - out[i];
}