#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Bprop2(const float* out, const float* layer1, float* dsyn2, const int count, const float alpha)
{
int i = blockDim.y*blockIdx.y + threadIdx.y; //256
int j = blockDim.x*blockIdx.x + threadIdx.x; //4
//int k = blockIdx.x;  //Data.count

atomicAdd(&dsyn2[i*4 + j], out[j] * layer1[256*(count) + i] * alpha);
}