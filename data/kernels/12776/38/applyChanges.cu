#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void applyChanges(float* syn, float* dsyn, int dim, float alpha)
{
int j = blockDim.x*blockIdx.x + threadIdx.x;
int k = blockDim.y*blockIdx.y + threadIdx.y;
syn[j*dim + k] += dsyn[j*dim + k] * alpha;
}