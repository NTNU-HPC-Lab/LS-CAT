#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardPass1(float* in, float* syn1, float* layer1)
{
int l = blockDim.x*blockIdx.x + threadIdx.x;
int j = blockDim.y*blockIdx.y + threadIdx.y;
int Y = 128;

atomicAdd(&layer1[l] , in[j] * syn1[j*Y + l]);

layer1[l] = 1.0/(1.0 + exp(layer1[l]));
}