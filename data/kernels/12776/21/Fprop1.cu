#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop1(const float* in, const float* syn1, float* layer1)
{
int i = threadIdx.x;                         //256
int j = blockDim.y*blockIdx.y + threadIdx.y; //64
int k = blockIdx.x;                          //Data.count
atomicAdd(&layer1[256*k + i], in[64*k + j] * syn1[j*256 + i]);
}