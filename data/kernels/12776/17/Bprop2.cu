#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Bprop2(const float* layer1, float* dsyn2, const float* out, const float alpha)
{
int i = threadIdx.x; //256
int j = blockDim.y*blockIdx.y + threadIdx.y; //10
int k = blockIdx.x;  //Data.count

atomicAdd(&dsyn2[i*10 + j], out[k*10 + j] * layer1[256*k + i] * alpha);
}