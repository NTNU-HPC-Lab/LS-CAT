#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardPass2(float* layer1, float* syn2, float* out)
{
int l = blockDim.x*blockIdx.x + threadIdx.x;
int Y = 128;
int Z = 10;

#pragma unroll
for (int j=0; j < Y; ++j)
out[l] += layer1[j] * syn2[j*Z + l];

out[l] = 1.0/(1.0 + exp(out[l]));
}