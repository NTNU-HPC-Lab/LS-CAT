#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float tanh_(float x)
{
// e**2x - 1
// ---------
// e**2x + 1
float exp2x =    exp(2.0*x);
return (exp2x - 1.0)/(exp2x + 1.0);
}
__global__ void LSTM1(float* layer1, float* lstm1, const float* gate1i, const float* gate1o, const int offset)
{
int i = blockDim.x*blockIdx.x + threadIdx.x; //256
float g_i = gate1i[256*offset + i];
float g_f = 1.0 - g_i;
float g_o = gate1o[256*offset + i];

float i_t = tanh_(layer1[256*offset + i]) * g_i;
float i_p = 0.0;
if (offset > 0)
i_p = g_f * lstm1[256*(offset-1) + i];
float sum = i_p + i_t;
lstm1[256*offset + i] = sum;
layer1[256*offset + i] = tanh_(sum) * g_o;
}