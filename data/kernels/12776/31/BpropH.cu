#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BpropH(const float* layer1, float* dlayer1, const float* synH, float* dsynH, const float alpha, const int offset)
{
int i = blockDim.x*blockIdx.x + threadIdx.x; //256
int j = blockDim.y*blockIdx.y + threadIdx.y; //256

atomicAdd(&dsynH[i*256 + j] , dlayer1[offset*256 + j] * layer1[(offset-1)*256 + i] * alpha);
atomicAdd(&dlayer1[(offset-1)*256 + i] , layer1[offset*256 + j] * synH[i*256 + j]);
}