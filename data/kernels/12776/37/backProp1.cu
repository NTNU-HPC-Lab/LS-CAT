#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backProp1(float* in, float* dsyn1, float* layer1, float* syn2, float* label, float* out)
{
int j = blockDim.x*blockIdx.x + threadIdx.x;
int k = blockDim.y*blockIdx.y + threadIdx.y;
float error = 0.0;

#pragma unroll
for (int l=0; l < 10; ++l)
error += (label[l] - out[l]) * syn2[k*10 + l];
float delta = error * (layer1[k]*(1-layer1[k]));
dsyn1[j*128 + k] += delta * in[j] / (60000.0/10.0);
}