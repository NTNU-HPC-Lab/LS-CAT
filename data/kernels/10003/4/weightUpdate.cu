#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")




using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void weightUpdate(float *d_W,float *d_D,float *d_N){
int2 pos;
pos.x = blockIdx.x*blockDim.x + threadIdx.x;//row j
pos.y = blockIdx.y*blockDim.y + threadIdx.y;//column k
int n = pos.x*blockDim.x*gridDim.y + pos.y;
float N = 0.1;
d_W[n] = d_W[n] + N*d_D[pos.y] * d_N[pos.x];
}