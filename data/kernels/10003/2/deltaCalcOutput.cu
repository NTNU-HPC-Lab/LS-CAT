#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")




using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void deltaCalcOutput(float *OutActivation, float *Outputdelta, float *targets){
int n = blockIdx.x*blockDim.x + threadIdx.x;
Outputdelta[n] = (targets[n] - OutActivation[n]) * (1 / (1 + exp(-OutActivation[n]))*(1 - 1 / (1 + exp(-OutActivation[n]))));
}