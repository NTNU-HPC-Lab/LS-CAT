#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")




using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void initWeights(float *dst, unsigned int seed){
//params are: seed,sequence num,offset,handle
int n = blockIdx.x*blockDim.x + threadIdx.x;
dst[n] = dst[n]/(float)(seed);
while(dst[n] > 5) {
dst[n]=dst[n]/2;
}
if (n%(seed % 3) == 0) {
dst[n] = dst[n] * -1;
}
}