#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")




using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void deltaCalcHidden(float *Activation,float *delta){
int n = blockIdx.x*blockDim.x + threadIdx.x;
delta[n] = delta[n] * (1 / (1 + exp(-Activation[n]))*(1 - 1 / (1 + exp(-Activation[n]))));
}