#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")




using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void printError(float *output,float *target) {
int n = blockIdx.x*blockDim.x + threadIdx.x;
float error = target[n] - output[n];
printf("%f \n", error );
}