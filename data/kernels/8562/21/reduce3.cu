#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce3(float *in, float *out, int size)
{
extern __shared__ float sdata[];
int tid = threadIdx.x;
int index = blockIdx.x * blockDim.x + threadIdx.x;
sdata[tid] = (index < size) ? in[index] : 0;

__syncthreads();
for(int s = blockDim.x/2; s>0; s>>=1)
{
if(tid<s)
sdata[tid] += sdata[tid + s];
__syncthreads();
}//end of for loop
if(tid == 0)
out[blockIdx.x] = sdata[0];
}//end of reduce3 kernal