#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( int *a, int dimx, int dimy )
{
int ix   = blockIdx.x*blockDim.x + threadIdx.x;
int iy   = blockIdx.y*blockDim.y + threadIdx.y;
int idx = iy*dimx + ix;

a[idx]  = a[idx]+1;
}