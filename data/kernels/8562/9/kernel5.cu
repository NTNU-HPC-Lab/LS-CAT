#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel5( int *a, int dimx, int dimy )
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * dimx + ix;
if(ix<dimx && iy < dimy)
a[idx] = blockIdx.y;
}