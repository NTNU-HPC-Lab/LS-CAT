#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply_simple(float *a, float *b, float *ab, size_t width)
{
//TODO: write the kernel to perform matrix a times b, store results into ab.
// width is the size of the square matrix along one dimension.
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if(row < width && col < width)
{
float pvalue = 0;
for(int k = 0; k < width; k++)
{
pvalue += a[row * width + k] * b[k * width +col];
}
ab[row * width + col] = pvalue;
}

}