#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel3( int *a, int dimx, int dimy )
{
int ix = blockIdx.x*blockDim.x + threadIdx.x;
int iy = blockIdx.y*blockDim.y + threadIdx.y;
int idx = iy*dimx + ix;
if(iy < dimy && ix < dimx)
a[idx] = idx;
}