#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel4( int *a, int dimx, int dimy )
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * dimx + ix;
if(ix<dimx && iy < dimy)
a[idx] = (threadIdx.y *  blockDim.x) + threadIdx.x;
}