#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize(int *values, int *max, float *output, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < n)
{
output[i] = (float)values[i] / (float)*max;
}
}