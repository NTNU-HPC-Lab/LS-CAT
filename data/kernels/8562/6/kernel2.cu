#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2( int *a, int dimx, int dimy )
{
int ix = blockIdx.x*blockDim.x + threadIdx.x;
int iy = blockIdx.y*blockDim.y + threadIdx.y;
int idx = iy * dimx + ix;
if(iy < dimy && ix < dimx)
a[idx] = (blockIdx.y * gridDim.x) + blockIdx.x;
}