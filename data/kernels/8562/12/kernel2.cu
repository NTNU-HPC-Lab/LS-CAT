#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2( int *a, int dimx, int dimy )
{
int ix = blockIdx.x*blockDim.x + threadIdx.x;
int iy = blockIdx.y*blockDim.y + threadIdx.y;
int idx = iy*dimx + ix;
a[idx] = (blockIdx.x + blockIdx.y);
}