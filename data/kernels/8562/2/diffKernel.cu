#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diffKernel( float *in, float *out, int n )
{
// Wrtie the kernel to implement the diff operation on an array
int id = (blockDim.x * blockIdx.x) + threadIdx.x;
if(id < n-1)
out[id] = in[id+1] - in[id];

}