#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width)
{
extern __shared__ float s_data[];
// TODO, implement this kernel below
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
y = y + 1; //because the edge of the data is not processed
// global thread(data) column index
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
x = x + 1; //because the edge of the data is not processed
if( y >=width - 1|| x >= width - 1 || y < 1 || x < 1 )// this is to check to make sure that the thread is within the array.
return;

int startRow = blockIdx.y;
int startCol = blockDim.x * blockIdx.x;

int s_rowwidth = blockDim.x +2; // because the blocks have to overlap on the right side that is why you add 2
int s_index0 = threadIdx.x +1; //row zero in s_data. you add one because you don't deal with the outer edge
int s_index1 = threadIdx.x + s_rowwidth + 1; //row one in s_data.so this goes to the other side
int s_index2 = threadIdx.x + 2 * s_rowwidth +1; //this is to get the last
//int s_index_result = threadIdx.x + 3 * s_rowwidth + 1;
int mid_row = blockIdx.x * blockDim.x + 1 + floatpitch * blockIdx.y;

int g_index0 = (mid_row -1) * floatpitch + startCol + 1+ threadIdx.x;
int g_index1 = (mid_row) * floatpitch + startCol  + 1 + threadIdx.x;
int g_index2 = (mid_row +1) * floatpitch +startCol + 1 + threadIdx.x;

if(startCol + startRow + 1 < width -1)
{
//copy the data from gobal mem to shared mem
s_data[s_index0] = g_dataA[g_index0];
s_data[s_index1] = g_dataA[g_index1];
s_data[s_index2] = g_dataA[g_index2];

}//end of if statement to populate the middle row of the current block
if(startRow == 0)
{
//copy the extra two columns in the globabl mem
s_data[s_index0 -1] = g_dataA[g_index0 - 1];
s_data[s_index1 -1] = g_dataA[g_index1 -1];
s_data[s_index2 -1] = g_dataA[g_index2 -1];
}//end of if statement to populate the edge row
if(threadIdx.x == width -3 - startCol || threadIdx.x == blockDim.x-1)
{
s_data[s_index0 + 1] = g_dataA[g_index0 +1];
s_data[s_index1 + 1] = g_dataA[g_index1 +1];
s_data[s_index2 +1] = g_dataA[g_index2 + 1];
}//end of if statement to populate the row below the middle row

__syncthreads();

//if( x >= width - 1|| y >= width - 1 || x < 1 || y < 1 )// this is to check to make sure that the thread is within the array.
//	return;

//this is copied from the other kernel
g_dataB[y * width + x] = (
0.2f * s_data[s_index1] +               //itself s_ind_1
0.1f * s_data[s_index0 -1] +       //N s_ind_0
0.1f * s_data[s_index0 +1] +       //NE s_ind_0
0.1f * s_data[s_index0   ] +       //E s_ind1
0.1f * s_data[s_index1 +1] +       //SE s_ind2
0.1f * s_data[s_index1 -1] +       //S s_ind2
0.1f * s_data[s_index2   ] +       //SW
0.1f * s_data[s_index2 -1] +       //W
0.1f * s_data[s_index2 +1]         //NW
) * 0.95f;//*/
}