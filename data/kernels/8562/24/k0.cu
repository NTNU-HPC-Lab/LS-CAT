#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k0( float* g_dataA, float* g_dataB, int pitch, int width )
{

// global thread(data) row index
unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
i = i + 1; //because the edge of the data is not processed

// global thread(data) column index
unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
j = j + 1; //because the edge of the data is not processed

// check the boundary
if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

g_dataB[i * pitch + j] = (
0.2f * g_dataA[i * pitch + j] +               //itself
0.1f * g_dataA[(i-1) * pitch +  j   ] +       //N
0.1f * g_dataA[(i-1) * pitch + (j+1)] +       //NE
0.1f * g_dataA[ i    * pitch + (j+1)] +       //E
0.1f * g_dataA[(i+1) * pitch + (j+1)] +       //SE
0.1f * g_dataA[(i+1) * pitch +  j   ] +       //S
0.1f * g_dataA[(i+1) * pitch + (j-1)] +       //SW
0.1f * g_dataA[ i    * pitch + (j-1)] +       //W
0.1f * g_dataA[(i-1) * pitch + (j-1)]         //NW
) * 0.95f;
}