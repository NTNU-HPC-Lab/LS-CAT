#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_max(int *values, int *max, int *reg_maxes, int num_regions, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int region = i % num_regions;
if(i < n)
{
int val = values[i];
if(atomicMax(&reg_maxes[region], val) < val)
{
atomicMax(max, val);
}//end of if statement
}//end of if i < n
}