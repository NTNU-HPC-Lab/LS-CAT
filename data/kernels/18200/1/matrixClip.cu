#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixClip(double *a, double min, double max, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){

if(a[y * cc + x] > max){
c[y * cc + x] = max;
}else{
if(a[y * cc + x] < min){
c[y * cc + x] = min;
}else{
c[y * cc + x] = a[y * cc + x];
}
}

}

}