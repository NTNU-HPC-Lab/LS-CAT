#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixLogit(double *a, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){
c[y * cc + x] = log(a[y * cc + x] / (1-a[y * cc + x]));
}

}