#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSigmoid(double *a, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){
c[y * cc + x] = 1.0/ (1+ exp(-a[y * cc + x]));
}

}