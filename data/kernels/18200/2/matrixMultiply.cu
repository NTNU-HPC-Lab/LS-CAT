#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(double *a, double *b, double *c, int cr, int cc, int ac, int bc){

long x = blockIdx.x * blockDim.x + threadIdx.x; // col
long y = blockIdx.y * blockDim.y + threadIdx.y; // row
double sum = 0;

if(x < cc && y < cr){

for(int k = 0; k<ac; k++){
sum+= a[y*ac+k] * b[k*bc+x];
}
c[y * cc + x] = sum;
}

}