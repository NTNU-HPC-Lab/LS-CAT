#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void detect_edges(unsigned char *input, unsigned char *output) {
int i = (blockIdx.x * 72) + threadIdx.x;
int x, y; // the pixel of interest
int b, d, f, h; // the pixels adjacent to the x,y used to calculate
int r; // the calculation result
y = i / width;;
x = i - (width * y);
if (x == 0 || y == 0 || x == width - 1 || y == height - 1) {
output[i] = 0;
} else {
b = i + width;
d = i - 1;
f = i + 1;
h = i - width;
r = (input[i] * 4) + (input[b] * -1) + (input[d] * -1) + (input[f] * -1)
+ (input[h] * -1);
if (r >= 0) {
output[i] = 0;
} else {
output[i] = 255;
}
}
}