#include "hip/hip_runtime.h"
#include "includes.h"

__device__ Query query_table(const int num_bucket, const int *bucket_start, const int key){
const unsigned int bucket_id = key;
const unsigned int list_start = (bucket_id > 0 ? bucket_start[bucket_id - 1] : 0);
const unsigned int next_list_start = bucket_start[bucket_id];
Query query(list_start, next_list_start);
return query;
}
__global__ void queryDevice(const int num_bucket, const int *bucket_start, const int key){
Query queryresult = query_table(num_bucket, bucket_start, key);
}