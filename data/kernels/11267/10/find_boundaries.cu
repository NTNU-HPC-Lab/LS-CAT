#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_boundaries(const int num_keys, const int num_bucket, const int *which_bucket, int *bucket_start){
int index = threadIdx.x + blockIdx.x*blockDim.x +blockIdx.y*blockDim.x*gridDim.x;
// Each thread looks at one entry in the sorted bucket index list
if (index >= num_keys){
return;
}
int previous_bucket = (index > 0 ? which_bucket[index - 1] : 0);
int my_bucket = which_bucket[index];
/*
*/
if (previous_bucket != my_bucket){
for (int i = previous_bucket; i < my_bucket; ++i){
bucket_start[i] = index;
}
}

/*
*/
if (index == num_keys - 1){
for (int i = my_bucket; i < num_bucket; ++i){
bucket_start[i] = num_keys;
}
}
}