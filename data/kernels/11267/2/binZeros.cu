#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binZeros(int *d_bin_count, int bin_size){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < bin_size){
d_bin_count[i] = 0;
}
}