#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeHessianListS2(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double gauss_d1, double *e_x_cov_x, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22, double *point_hessians, double *hessians, int valid_voxel_num)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int row = blockIdx.y;
int col = blockIdx.z;

if (row < 6 && col < 6) {
double *h = hessians + (row * 6 + col) * valid_points_num;
double *tmp_ph0 = point_hessians + ((3 * row) * 6 + col) * valid_points_num;
double *tmp_ph1 = tmp_ph0 + 6 * valid_points_num;
double *tmp_ph2 = tmp_ph1 + 6 * valid_points_num;

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];
double d_x = static_cast<double>(trans_x[pid]);
double d_y = static_cast<double>(trans_y[pid]);
double d_z = static_cast<double>(trans_z[pid]);
double ph0 = tmp_ph0[i];
double ph1 = tmp_ph1[i];
double ph2 = tmp_ph2[i];

double final_hessian = h[i];

for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
//Transformed coordinates
int vid = voxel_id[j];
double tmp_ex = e_x_cov_x[j];

if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
tmp_ex *= gauss_d1;

final_hessian += (d_x - centroid_x[vid]) * (icov00[vid] * ph0 + icov01[vid] * ph1 + icov02[vid] * ph2) * tmp_ex;
final_hessian += (d_y - centroid_y[vid]) * (icov10[vid] * ph0 + icov11[vid] * ph1 + icov12[vid] * ph2) * tmp_ex;
final_hessian += (d_z - centroid_z[vid]) * (icov20[vid] * ph0 + icov21[vid] * ph1 + icov22[vid] * ph2) * tmp_ex;

}
}

h[i] = final_hessian;
}
}
}