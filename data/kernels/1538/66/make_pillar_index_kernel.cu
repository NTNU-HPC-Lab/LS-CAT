#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_pillar_index_kernel( int* dev_pillar_count_histo, int* dev_counter, int* dev_pillar_count, int* dev_x_coors, int* dev_y_coors, float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, float* dev_num_points_per_pillar, int* dev_sparse_pillar_map, const int max_pillars, const int max_points_per_pillar, const int GRID_X_SIZE, const float PILLAR_X_SIZE, const float PILLAR_Y_SIZE, const int NUM_INDS_FOR_SCAN)
{
int x = blockIdx.x;
int y = threadIdx.x;
int num_points_at_this_pillar = dev_pillar_count_histo[y*GRID_X_SIZE + x];
if(num_points_at_this_pillar == 0)
{
return;
}

int count = atomicAdd(dev_counter, 1);
if(count < max_pillars)
{
atomicAdd(dev_pillar_count, 1);
if(num_points_at_this_pillar >= max_points_per_pillar)
{
dev_num_points_per_pillar[count] = max_points_per_pillar;
}
else
{
dev_num_points_per_pillar[count] = num_points_at_this_pillar;
}
dev_x_coors[count] = x;
dev_y_coors[count] = y;

//TODO Need to be modified after making properly trained weight
// Will be modified in ver 1.1
// x_offset = self.vx / 2 + pc_range[0]
// y_offset = self.vy / 2 + pc_range[1]
// x_sub = coors_x.unsqueeze(1) * 0.16 + x_offset
// y_sub = coors_y.unsqueeze(1) * 0.16 + y_offset
dev_x_coors_for_sub[count] =  x*  PILLAR_X_SIZE + 0.1f;
dev_y_coors_for_sub[count] =  y*  PILLAR_Y_SIZE + -39.9f;
dev_sparse_pillar_map[y*NUM_INDS_FOR_SCAN + x] = 1;
}
}