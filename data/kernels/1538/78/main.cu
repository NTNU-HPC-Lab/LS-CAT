#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computePointGradients1.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
float *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
float *z = NULL;
hipMalloc(&z, XSIZE*YSIZE);
int points_num = 1;
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int valid_points_num = 1;
double *dj_ang = NULL;
hipMalloc(&dj_ang, XSIZE*YSIZE);
double *pg24 = NULL;
hipMalloc(&pg24, XSIZE*YSIZE);
double *pg05 = NULL;
hipMalloc(&pg05, XSIZE*YSIZE);
double *pg15 = NULL;
hipMalloc(&pg15, XSIZE*YSIZE);
double *pg25 = NULL;
hipMalloc(&pg25, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computePointGradients1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dj_ang,pg24,pg05,pg15,pg25);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computePointGradients1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dj_ang,pg24,pg05,pg15,pg25);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computePointGradients1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dj_ang,pg24,pg05,pg15,pg25);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}