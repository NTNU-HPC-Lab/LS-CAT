#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointHessian0(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph93, double *ph103, double *ph113, double *ph123, double *ph94, double *ph133, double *ph104, double *ph143, double *ph114, double *ph153, double *ph95, double *ph163, double *ph105, double *ph173, double *ph115)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
__shared__ double h_ang[18];

if (threadIdx.x < 18) {
h_ang[threadIdx.x] = dh_ang[threadIdx.x];
}

__syncthreads();

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];

//Orignal coordinates
double o_x = static_cast<double>(x[pid]);
double o_y = static_cast<double>(y[pid]);
double o_z = static_cast<double>(z[pid]);


ph93[i] = 0;
ph103[i] = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
ph113[i] = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];

ph123[i] = ph94[i] = 0;
ph133[i] = ph104[i] = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];
ph143[i] = ph114[i] = o_x * h_ang[9] + o_y * h_ang[10] + o_z * h_ang[11];

ph153[i] = ph95[i] = 0;
ph163[i] = ph105[i] = o_x * h_ang[12] + o_y * h_ang[13] + o_z * h_ang[14];
ph173[i] = ph115[i] = o_x * h_ang[15] + o_y * h_ang[16] + o_z * h_ang[17];

}
}