#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000


__global__ void vec_sum(float* a, float* b, float* c) {
int bid = blockIdx.x;
if (bid < N) {
c[bid] = a[bid] + b[bid];
}
}