#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SegmentAllocLocInit(ushort2* gSegments, const uint32_t segmentCount)
{
unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
if(globalId >= segmentCount) return;
gSegments[globalId].x = 0xFFFF;
gSegments[globalId].y = 0xFFFF;
}