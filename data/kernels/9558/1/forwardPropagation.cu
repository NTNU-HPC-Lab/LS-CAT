#include "hip/hip_runtime.h"
#include "includes.h"






__global__ void forwardPropagation(float *a_d , float *b_d ,int size)
{
int idx = threadIdx.x;
int idy = threadIdx.y;

__shared__ float temp[16][16];

temp[idy][idx] = a_d[(idy * (size+1)) + idx] ;

for(int i = 1 ; i < size; i++)
{
if((idy + i) < size)
{
float var1 =(-1)*(temp[i-1][i-1] / temp[i+idy][i-1]);
temp[i+idy][idx] = temp[i-1][idx] + ((var1) * (temp[i+idy][idx]));
}
__syncthreads();
}

b_d[idy*(size+1) + idx] = temp[idy][idx];
}