#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void gpuSummationReduce(float *in, float *out, int n)
{
extern __shared__ float sdata[];

// load shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

sdata[tid] = (i < n) ? in[i] : 0;

__syncthreads();

// do reduction in shared mem
for (unsigned int s=1; s < blockDim.x; s *= 2)
{
// modulo arithmetic is slow!
if ((tid % (2*s)) == 0)
{

sdata[tid] += sdata[tid + s]; //bigger number stored in low index
}

__syncthreads();
}

// write result for this block to global mem
if (tid == 0) out[blockIdx.x] = sdata[0];
}