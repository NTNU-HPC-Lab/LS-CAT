#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

#define MAX_N_TERMS 10



__global__ void MC_Integratev1(float* degrees,int dimension,int n_terms,float* I_val,hiprandState *states, long int seed,int thread_max_iterations)
{
//Get the Global ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

float x;
float I = 0.0;
float f[MAX_N_TERMS];
//float* f =new float[n_terms];

//Initialize the random number generator
hiprand_init(seed, id, 0, &states[id]);

for (int iter_count=0;iter_count< thread_max_iterations;iter_count++)
{
//Initialize f with the coefficients
for (int term_i=0;term_i<n_terms;term_i++)
{
f[term_i]=degrees[(2+term_i)*dimension];
}

for (int d=1;d<dimension;d++)
{
//Generate a random number in the range of the limits of this dimension
x = hiprand_uniform (&states[id]);    //x between 0 and 1
//Generate dimension sample based on the limits of the dimension
x = x*(degrees[1*dimension+d]-degrees[0*dimension+d])+degrees[0*dimension+d];
for (int term_i=0;term_i<n_terms;term_i++)
{
//Multiply f of this term by x^(power of this dimension in this term)
f[term_i]*=pow(x,degrees[(2+term_i)*dimension+d]);
}
}
//Add the evaluation to the private summation
for (int term_i=0;term_i<n_terms;term_i++)
{
I+=f[term_i];
}
}
//Add the private summation to the global summation
atomicAdd(I_val,I);

}