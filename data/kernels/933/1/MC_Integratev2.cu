#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

#define MAX_N_TERMS 10



__global__ void MC_Integratev2(float* degrees_g,int dimension,int n_terms,float* I_val, long int seed,int thread_max_iterations)
{
//Get the global and local ids
int id = blockIdx.x*blockDim.x+threadIdx.x;
int lid=threadIdx.x;
float x;
float I = 0.0;
float f[MAX_N_TERMS];
//float* f =new float[n_terms];

//Dynamically allocate shared memory for 'degrees' and 'I_shared'
extern __shared__ float shared_mem[];
float* I_shared = shared_mem;
I_shared[0]=0;
float* degrees = &shared_mem[1];

//Initialize the local copy of 'degrees' for the shared copy
if(lid<(2+n_terms)*dimension)
{
//copy one element of degrees
degrees[lid]=degrees_g[lid];
}

// Create a state in private memory
hiprandState state;
//Initialize the random number generator
hiprand_init(seed,id,0,&state);

//Synchronize all threads to assure that 'degrees' is initialized
__syncthreads();

for (int iter_count=0;iter_count< thread_max_iterations;iter_count++)
{
//Initialize f with the coefficients
for (int term_i=0;term_i<n_terms;term_i++)
{
f[term_i]=degrees[(2+term_i)*dimension];
}

for (int d=1;d<dimension;d++)
{
//Generate a random number in the range of the limits of this dimension
x = hiprand_uniform (&state);    //x between 0 and 1
//Generate dimension sample based on the limits of the dimension
x = x*(degrees[1*dimension+d]-degrees[0*dimension+d])+degrees[0*dimension+d];
for (int term_i=0;term_i<n_terms;term_i++)
{
//Multiply f of this term by x^(power of this dimension in this term)
f[term_i]*=pow(x,degrees[(2+term_i)*dimension+d]);
}

}
//Add the evaluation to the private summation
for (int term_i=0;term_i<n_terms;term_i++)
{
I+=f[term_i];
}
}
//Add the private summation to the shared summation
atomicAdd(I_shared,I);
//Synchronize all the threads to assure they all added their private summations to the shared summation
__syncthreads();
//Thread 0 in the block add the shared summation to the global summation
if(lid==0)
{
atomicAdd(I_val,*I_shared);
}


}