#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void hook_init(int* parent, Edge* edge_list, int e){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
long long int x;
int u, v, mx, mn;
if(id < e){
x = edge_list[id].x;
v = (int) x & 0xFFFFFFFF;
u = (int) (x >> 32);

mx = max(u, v);
mn = u + v - mx;
parent[mx] = mn;
}
return;
}