#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void initialize_active_edges(bool* active_edges, int e){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
if(id < e)
active_edges[id] = true;
return;
}