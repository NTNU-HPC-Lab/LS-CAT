#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void update_states(int* parent, int* vertex_state, int n){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
if(id < n)
vertex_state[id] = parent[id] == id ? 0 : 1;
return;
}