#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void initialize_parent(int* parent, int n){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
if(id < n)
parent[id] = id;
return;
}