#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void leaf_pointer_jumping(int* parent, int* vertex_state, int n){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
int parent_id, grandparent_id;
if(id < n)
if(vertex_state[id] == 1){
parent_id = parent[id];
grandparent_id = parent[parent_id];
parent[id] = grandparent_id;
}
return;
}