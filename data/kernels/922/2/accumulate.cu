#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void accumulate(Edge* edge_list, bool* cross_edges, int* indices, int e){
int bid = blockIdx.x;
int id = bid*blockDim.x + threadIdx.x;
Edge temp;
temp.x = 0;
if(id < e)
if(cross_edges[id])
temp = edge_list[id];
__syncthreads();
if(temp.x)
edge_list[indices[id]] = temp;
return;
}