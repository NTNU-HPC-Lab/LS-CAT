#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void process_cross_edges(int* parent, Edge* edge_list, int e, bool* flag, bool* cross_edges){
int bid = blockIdx.x;
int tid = threadIdx.x;
int id = bid*blockDim.x + tid;
long long int x;
int u, v, mn, mx, parent_u, parent_v;
__shared__ bool block_flag;
if(tid == 0)
block_flag = false;
__syncthreads();
if(id < e)
if(cross_edges[id]){
x = edge_list[id].x;
v = (int) x & 0xFFFFFFFF;
u = (int) (x >> 32);

parent_u = parent[u];
parent_v = parent[v];

mn = min(parent_u, parent_v);
mx = parent_u + parent_v - mn;

if(parent_u == parent_v)
cross_edges[id] = false;
else{
parent[mx] = mn;
block_flag = true;
}
}
__syncthreads();

if(tid == 0)
if(block_flag)
*flag = true;
return;
}