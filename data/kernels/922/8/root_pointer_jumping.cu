#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void root_pointer_jumping(int* parent, int* vertex_state, int n, bool* flag){
int bid = blockIdx.x;
int tid = threadIdx.x;
int id = bid*blockDim.x + tid;
int parent_id, grandparent_id;
__shared__ bool block_flag;
if(tid == 0)
block_flag = false;
__syncthreads();
if(id < n)
if(vertex_state[id] == 0){
parent_id = parent[id];
grandparent_id = parent[parent_id];
if(parent_id != grandparent_id){
parent[id] = grandparent_id;
block_flag = true;
}
else
vertex_state[id] = -1;
}
if(tid == 0)
if(block_flag)
*flag = true;
return;
}