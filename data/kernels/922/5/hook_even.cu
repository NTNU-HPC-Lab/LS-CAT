#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};



///*
//*/










__global__ void hook_even(int* parent, Edge* edge_list, int e, bool* flag, bool* active_edges){
int bid = blockIdx.x;
int tid = threadIdx.x;
int id = bid*blockDim.x + tid;
long long int x;
int u, v, mx, mn, parent_u, parent_v;
__shared__ bool block_flag;
if(tid == 0)
block_flag = false;
__syncthreads();
if(id < e)
if(active_edges[id]){
x = edge_list[id].x;
v = (int) x & 0xFFFFFFFF;
u = (int) (x >> 32);

parent_u = parent[u];
parent_v = parent[v];

mx = max(parent_u, parent_v);
mn = parent_u + parent_v - mx;

if(parent_u == parent_v)
active_edges[id] = false;
else{
parent[mn] = mx;
block_flag = true;
}
}
__syncthreads();

if(tid == 0)
if(block_flag)
*flag = true;
return;
}