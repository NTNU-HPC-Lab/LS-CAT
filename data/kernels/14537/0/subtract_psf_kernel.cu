#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtract_psf_kernel( double * res_p_trans , const double * psf_p_trans , const int stopx , const int stopy , const int diff , const int linsize , const double peak_x_gain ) {
const int
x =  threadIdx.x + (blockIdx.x * blockDim.x)
, y =  threadIdx.y + (blockIdx.y * blockDim.y)
, tid = y * linsize + x
;
if (x < stopx && y < stopy) res_p_trans[tid] -= peak_x_gain * psf_p_trans[tid + diff];
}