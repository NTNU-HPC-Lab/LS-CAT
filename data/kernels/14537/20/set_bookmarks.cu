#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_bookmarks(int2* vis_in, int npts, int blocksize, int blockgrid, int* bookmarks) {
for (int q=threadIdx.x+blockIdx.x*blockDim.x;q<=npts;q+=gridDim.x*blockDim.x) {
int2 this_vis = vis_in[q];
int2 last_vis = vis_in[q-1];
int main_x = this_vis.x/GCF_GRID/blocksize;
int main_x_last = last_vis.x/GCF_GRID/blocksize;
int main_y = this_vis.y/GCF_GRID/blocksize;
int main_y_last = last_vis.y/GCF_GRID/blocksize;
if (0==q) {
main_y_last=0;
main_x_last=-1;
}
if (npts==q) main_x = main_y = blockgrid;
if (main_x != main_x_last || main_y != main_y_last)  {
for (int z=main_y_last*blockgrid+main_x_last+1;
z<=main_y*blockgrid+main_x; z++) {
bookmarks[z] = q;
}
}
}
}