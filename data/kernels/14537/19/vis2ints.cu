#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vis2ints(double scale, double2 *vis_in, int2* vis_out, int npts) {
for (int q=threadIdx.x+blockIdx.x*blockDim.x;
q<npts;
q+=gridDim.x*blockDim.x) {
double2 inn = vis_in[q];
inn.x *= scale;
inn.y *= scale;
int main_y = floor(inn.y);
int sub_y = floor(GCF_GRID*(inn.y-main_y));
int main_x = floor(inn.x);
int sub_x = floor(GCF_GRID*(inn.x-main_x));
vis_out[q].x = main_x*GCF_GRID+sub_x;
vis_out[q].y = main_y*GCF_GRID+sub_y;
}
}