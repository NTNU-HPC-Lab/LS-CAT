#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_kernel_multi_block_spill( volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_local_mem)
{

extern volatile __shared__ unsigned int b_q[];

volatile __shared__ unsigned int b_q_length[1];
volatile __shared__ unsigned int b_offset[1];
//get the threadId
unsigned int tid=threadIdx.x + blockDim.x * blockIdx.x;
unsigned int lid=threadIdx.x;

//initialize the block queue length and warp queue offset
if (lid == 0 )
{
b_q_length[0]=0;
b_offset[0]=0;
}

__syncthreads();
//Initialize the warp queue sizes to 0
if(tid<frontier_len)
{
//get the nodes to traverse from block queue
unsigned int node_to_process=frontier[tid];
visited[node_to_process]=0;
//get the offsets of the vertex in the edge list
unsigned int offset=edgeArray[node_to_process];
unsigned int next=edgeArray[node_to_process+1];

//Iterate through the neighbors of the vertex
while(offset<next)
{
//get neighbor
unsigned int nid=edgeArrayAux[offset];
//get its cost
unsigned int v=atomicMin((unsigned int *)&cost[nid],
cost[node_to_process]+1);
//if cost is less than previously set add to frontier
if(v>cost[node_to_process]+1)
{
int is_in_frontier=atomicExch((int *)&visited[nid],1);
//if node already in frontier do nothing
if(is_in_frontier==0)
{
//increment the warp queue size
unsigned int t=atomicAdd((unsigned int *)&b_q_length[0],
1);
if(t<max_local_mem)
{
b_q[t]=nid;
}
//write to global memory if shared memory full
else
{
int off=atomicAdd((unsigned int *)frontier_length,
1);
frontier2[off]=nid;
}
}
}
offset++;
}
}

__syncthreads();

//get block queue offset in global queue
if(lid==0)
{
if(b_q_length[0] > max_local_mem)
{
b_q_length[0]=max_local_mem;
}
b_offset[0]=atomicAdd((unsigned int *)frontier_length,b_q_length[0]);
}
__syncthreads();

//copy block queue to frontier
if(lid < b_q_length[0])
frontier2[lid+b_offset[0]]=b_q[lid];
}