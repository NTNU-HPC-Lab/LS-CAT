#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelInitNablaW(float *nabla_w,int tws) {
if ((blockIdx.x*blockDim.x+threadIdx.x)<tws) {
nabla_w[blockIdx.x*blockDim.x+threadIdx.x]=0.0;
}
}