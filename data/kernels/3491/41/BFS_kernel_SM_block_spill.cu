#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void __gpu_sync(int blocks_to_synch)
{
__syncthreads();
//thread ID in a block
int tid_in_block= threadIdx.x;


// only thread 0 is used for synchronization
if (tid_in_block == 0)
{
atomicAdd((int *)&g_mutex, 1);
//only when all blocks add 1 to g_mutex will
//g_mutex equal to blocks_to_synch
while(g_mutex < blocks_to_synch);
}
__syncthreads();
}
__global__ void BFS_kernel_SM_block_spill( volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_local_mem)
{
extern volatile __shared__ unsigned int b_q[];

volatile __shared__ unsigned int b_q_length[1];
volatile __shared__ unsigned int b_offset[1];

//get the threadId
unsigned int tid=threadIdx.x + blockDim.x * blockIdx.x;
unsigned int lid=threadIdx.x;

int loop_index=0;
unsigned int l_mutex=g_mutex2;
unsigned int f_len=frontier_len;
while(1)
{
//initialize the block queue length and warp queue offset
if (lid==0)
{
b_q_length[0]=0;
b_offset[0]=0;
}
__syncthreads();
//Initialize the warp queue sizes to 0
if(tid<f_len)
{
//get the nodes to traverse from block queue
unsigned int node_to_process;

if(loop_index==0)
node_to_process=frontier[tid];
else
node_to_process=frontier2[tid];

//remove from frontier
visited[node_to_process]=0;
//get the offsets of the vertex in the edge list
unsigned int offset=edgeArray[node_to_process];
unsigned int next=edgeArray[node_to_process+1];

//Iterate through the neighbors of the vertex
while(offset<next)
{
//get neighbor
unsigned int nid=edgeArrayAux[offset];
//get its cost
unsigned int v=atomicMin((unsigned int *)&cost[nid],
cost[node_to_process]+1);
//if cost is less than previously set add to frontier
if(v>cost[node_to_process]+1)
{
int is_in_frontier=atomicExch((int *)&visited[nid],1);
//if node already in frontier do nothing
if(is_in_frontier==0)
{
//increment the warp queue size
unsigned int t=atomicAdd((unsigned int *)&b_q_length[0],
1);
if(t<max_local_mem)
{
b_q[t]=nid;
}
//write to global memory if shared memory full
else
{
int off=atomicAdd((unsigned int *)g_q_offsets,1);
if(loop_index==0)
frontier2[off]=nid;
else
frontier[off]=nid;
}
}
}
offset++;
}
}
//get offset of block queue in global queue
__syncthreads();
if(lid==0)
{
if(b_q_length[0] > max_local_mem)
{
b_q_length[0] = max_local_mem;
}
b_offset[0]=atomicAdd((unsigned int *)g_q_offsets,b_q_length[0]);
}
__syncthreads();

l_mutex+=gridDim.x;
__gpu_sync(l_mutex);

//store frontier size
if(tid==0)
{
g_q_size[0]=g_q_offsets[0];
g_q_offsets[0]=0;
}

//copy block queue to global queue
if(lid < b_q_length[0])
{
if(loop_index==0)
frontier2[lid+b_offset[0]]=b_q[lid];
else
frontier[lid+b_offset[0]]=b_q[lid];
}

l_mutex+=gridDim.x;
__gpu_sync(l_mutex);

//if frontier exceeds SM blocks or less than 1 block exit
if(g_q_size[0] < blockDim.x ||
g_q_size[0] > blockDim.x * gridDim.x)
{

//TODO:Call the 1-block bfs right here
break;
}
loop_index=(loop_index+1)%2;
//store the current frontier size
f_len=g_q_size[0];
}

if(loop_index==0)
{
for(int i=tid;i<g_q_size[0];i += blockDim.x*gridDim.x)
frontier[i]=frontier2[i];
}

if(tid==0)
{
frontier_length[0]=g_q_size[0];
}
}