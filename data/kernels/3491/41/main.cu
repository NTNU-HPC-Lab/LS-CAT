#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "BFS_kernel_SM_block_spill.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
volatile unsigned int *frontier = NULL;
hipMalloc(&frontier, XSIZE*YSIZE);
volatile unsigned int *frontier2 = NULL;
hipMalloc(&frontier2, XSIZE*YSIZE);
unsigned int frontier_len = 1;
volatile unsigned int *cost = NULL;
hipMalloc(&cost, XSIZE*YSIZE);
volatile int *visited = NULL;
hipMalloc(&visited, XSIZE*YSIZE);
unsigned int *edgeArray = NULL;
hipMalloc(&edgeArray, XSIZE*YSIZE);
unsigned int *edgeArrayAux = NULL;
hipMalloc(&edgeArrayAux, XSIZE*YSIZE);
unsigned int numVertices = 1;
unsigned int numEdges = 1;
volatile unsigned int *frontier_length = NULL;
hipMalloc(&frontier_length, XSIZE*YSIZE);
const unsigned int max_local_mem = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
BFS_kernel_SM_block_spill<<<gridBlock,threadBlock>>>(frontier,frontier2,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,max_local_mem);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
BFS_kernel_SM_block_spill<<<gridBlock,threadBlock>>>(frontier,frontier2,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,max_local_mem);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
BFS_kernel_SM_block_spill<<<gridBlock,threadBlock>>>(frontier,frontier2,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,max_local_mem);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}