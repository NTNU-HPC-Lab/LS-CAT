#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void triad(float* A, float* B, float* C, float s)
{
int gid = threadIdx.x + (blockIdx.x * blockDim.x);
C[gid] = A[gid] + s*B[gid];
}