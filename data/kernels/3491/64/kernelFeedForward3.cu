#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward3(float *zs,float *biases,int b_off,float *activations) {
zs[(blockIdx.x*blockDim.x)+threadIdx.x]+=biases[b_off+threadIdx.x];
activations[(blockIdx.x*blockDim.x)+threadIdx.x]=1.0/(1.0+expf(-zs[(blockIdx.x*blockDim.x)+threadIdx.x]));
}