#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward2(float *zs,float *biases,int b_off,float *activations) {
zs[threadIdx.x]+=biases[b_off+threadIdx.x];
activations[threadIdx.x]=1.0/(1.0+expf(-zs[threadIdx.x]));
}