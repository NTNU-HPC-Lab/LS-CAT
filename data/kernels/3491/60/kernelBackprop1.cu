#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop1(float *delta_nabla_w,int w_off,float *activations,float *delta_nabla_b,int b_off) {
delta_nabla_w[w_off+(blockIdx.x*blockDim.x)+threadIdx.x]=activations[threadIdx.x]*delta_nabla_b[b_off+blockIdx.x];
//delta_nabla_w[w_off+(threadIdx.x*gridDim.x)+blockIdx.x]=activations[threadIdx.x]*delta_nabla_b[b_off+blockIdx.x];
}