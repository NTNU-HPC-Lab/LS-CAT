#include "hip/hip_runtime.h"
#include "includes.h"
__device__ uint scanLocalMem(const uint val, uint* s_data)
{
// Shared mem is 512 uints long, set first half to 0
int idx = threadIdx.x;
s_data[idx] = 0.0f;
__syncthreads();

// Set 2nd half to thread local sum (sum of the 4 elems from global mem)
idx += blockDim.x; // += 256

// Some of these __sync's are unnecessary due to warp synchronous
// execution.  Right now these are left in to be consistent with
// opencl version, since that has to execute on platforms where
// thread groups are not synchronous (i.e. CPUs)
uint t;
s_data[idx] = val;     __syncthreads();
t = s_data[idx -  1];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx -  2];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx -  4];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx -  8];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx - 16];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx - 32];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx - 64];  __syncthreads();
s_data[idx] += t;      __syncthreads();
t = s_data[idx - 128]; __syncthreads();
s_data[idx] += t;      __syncthreads();

return s_data[idx-1];
}
__global__ void scan(uint *g_odata, uint* g_idata, uint* g_blockSums, const int n, const bool fullBlock, const bool storeSum)
{
__shared__ uint s_data[512];

// Load data into shared mem
uint4 tempData;
uint4 threadScanT;
uint res;
uint4* inData  = (uint4*) g_idata;

const int gid = (blockIdx.x * blockDim.x) + threadIdx.x;
const int tid = threadIdx.x;
const int i = gid * 4;

// If possible, read from global mem in a uint4 chunk
if (fullBlock || i + 3 < n)
{
// scan the 4 elems read in from global
tempData       = inData[gid];
threadScanT.x = tempData.x;
threadScanT.y = tempData.y + threadScanT.x;
threadScanT.z = tempData.z + threadScanT.y;
threadScanT.w = tempData.w + threadScanT.z;
res = threadScanT.w;
}
else
{   // if not, read individual uints, scan & store in lmem
threadScanT.x = (i < n) ? g_idata[i] : 0.0f;
threadScanT.y = ((i+1 < n) ? g_idata[i+1] : 0.0f) + threadScanT.x;
threadScanT.z = ((i+2 < n) ? g_idata[i+2] : 0.0f) + threadScanT.y;
threadScanT.w = ((i+3 < n) ? g_idata[i+3] : 0.0f) + threadScanT.z;
res = threadScanT.w;
}

res = scanLocalMem(res, s_data);
__syncthreads();

// If we have to store the sum for the block, have the last work item
// in the block write it out
if (storeSum && tid == blockDim.x-1) {
g_blockSums[blockIdx.x] = res + threadScanT.w;
}

// write results to global memory
uint4* outData = (uint4*) g_odata;

tempData.x = res;
tempData.y = res + threadScanT.x;
tempData.z = res + threadScanT.y;
tempData.w = res + threadScanT.z;

if (fullBlock || i + 3 < n)
{
outData[gid] = tempData;
}
else
{
if ( i    < n) { g_odata[i]   = tempData.x;
if ((i+1) < n) { g_odata[i+1] = tempData.y;
if ((i+2) < n) { g_odata[i+2] = tempData.z; } } }
}
}