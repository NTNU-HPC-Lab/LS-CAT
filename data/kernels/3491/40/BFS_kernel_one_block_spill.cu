#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_kernel_one_block_spill( volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_local_mem)
{

extern volatile __shared__ unsigned int s_mem[];

//block queues
unsigned int *b_q=(unsigned int *)&s_mem[0];
unsigned int *b_q2=(unsigned int *)&s_mem[max_local_mem];

volatile __shared__ unsigned int b_offset[1];
volatile __shared__ unsigned int b_q_length[1];
//get the threadId
unsigned int tid=threadIdx.x;
//copy frontier queue from global queue to local block queue
if(tid<frontier_len)
{
b_q[tid]=frontier[tid];
}

unsigned int f_len=frontier_len;
while(1)
{
//Initialize the block queue size to 0
if(tid==0)
{
b_q_length[0]=0;
b_offset[0]=0;
}
__syncthreads();
if(tid<f_len)
{
//get the nodes to traverse from block queue
unsigned int node_to_process=*(volatile unsigned int *)&b_q[tid];
//remove from frontier
visited[node_to_process]=0;
//get the offsets of the vertex in the edge list
unsigned int offset = edgeArray[node_to_process];
unsigned int next   = edgeArray[node_to_process+1];

//Iterate through the neighbors of the vertex
while(offset<next)
{
//get neighbor
unsigned int nid=edgeArrayAux[offset];
//get its cost
unsigned int v=atomicMin((unsigned int *)&cost[nid],
cost[node_to_process]+1);
//if cost is less than previously set add to frontier
if(v>cost[node_to_process]+1)
{
int is_in_frontier=atomicExch((int *)&visited[nid],1);
//if node already in frontier do nothing
if(is_in_frontier==0)
{
//increment the warp queue size
unsigned int t=
atomicAdd((unsigned int *)&b_q_length[0],1);
if(t< max_local_mem)
{
b_q2[t]=nid;
}
//write to global memory if shared memory full
else
{
int off=atomicAdd((unsigned int *)&b_offset[0],
1);
frontier[off]=nid;
}
}
}
offset++;
}
}
__syncthreads();

if(tid<max_local_mem)
b_q[tid]=*(volatile unsigned int *)&b_q2[tid];

__syncthreads();
//Traversal complete exit
if(b_q_length[0]==0)
{
if(tid==0)
frontier_length[0]=0;
return;
}
// If frontier exceeds one block in size copy warp queues to
//global frontier queue and exit
else if( b_q_length[0] > blockDim.x || b_q_length[0] > max_local_mem)
{
if(tid<(b_q_length[0]-b_offset[0]))
frontier[b_offset[0]+tid]= *(volatile unsigned int *)&b_q[tid];
if(tid==0)
{
frontier_length[0] = b_q_length[0];
}
return;
}
f_len=b_q_length[0];
__syncthreads();
}
}