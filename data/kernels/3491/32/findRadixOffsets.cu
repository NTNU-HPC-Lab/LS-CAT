#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findRadixOffsets(uint2* keys, uint* counters, uint* blockOffsets, uint startbit, uint numElements, uint totalBlocks)
{
__shared__ uint  sStartPointers[16];
extern __shared__ uint sRadix1[];

uint groupId = blockIdx.x;
uint localId = threadIdx.x;
uint groupSize = blockDim.x;

uint2 radix2;
radix2 = keys[threadIdx.x + (blockIdx.x * blockDim.x)];

sRadix1[2 * localId]     = (radix2.x >> startbit) & 0xF;
sRadix1[2 * localId + 1] = (radix2.y >> startbit) & 0xF;

// Finds the position where the sRadix1 entries differ and stores start
// index for each radix.
if(localId < 16)
{
sStartPointers[localId] = 0;
}
__syncthreads();

if((localId > 0) && (sRadix1[localId] != sRadix1[localId - 1]) )
{
sStartPointers[sRadix1[localId]] = localId;
}
if(sRadix1[localId + groupSize] != sRadix1[localId + groupSize - 1])
{
sStartPointers[sRadix1[localId + groupSize]] = localId + groupSize;
}
__syncthreads();

if(localId < 16)
{
blockOffsets[groupId*16 + localId] = sStartPointers[localId];
}
__syncthreads();

// Compute the sizes of each block.
if((localId > 0) && (sRadix1[localId] != sRadix1[localId - 1]) )
{
sStartPointers[sRadix1[localId - 1]] =
localId - sStartPointers[sRadix1[localId - 1]];
}
if(sRadix1[localId + groupSize] != sRadix1[localId + groupSize - 1] )
{
sStartPointers[sRadix1[localId + groupSize - 1]] =
localId + groupSize - sStartPointers[sRadix1[localId +
groupSize - 1]];
}

if(localId == groupSize - 1)
{
sStartPointers[sRadix1[2 * groupSize - 1]] =
2 * groupSize - sStartPointers[sRadix1[2 * groupSize - 1]];
}
__syncthreads();

if(localId < 16)
{
counters[localId * totalBlocks + groupId] = sStartPointers[localId];
}
}