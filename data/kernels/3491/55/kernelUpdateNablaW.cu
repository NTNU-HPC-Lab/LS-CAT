#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateNablaW(float *nabla_w,float *delta_nabla_w,int tws) {
if ((blockIdx.x*blockDim.x+threadIdx.x)<tws) {
nabla_w[blockIdx.x*blockDim.x+threadIdx.x]+=delta_nabla_w[blockIdx.x*blockDim.x+threadIdx.x];
}
}