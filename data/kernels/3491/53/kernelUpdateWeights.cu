#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateWeights(float *nabla_w,float *weights,int tws,float eta,float mini_batch_size) {

float rate=eta/mini_batch_size;

if ((blockIdx.x*blockDim.x+threadIdx.x)<tws) {
weights[blockIdx.x*blockDim.x+threadIdx.x]-=rate*nabla_w[blockIdx.x*blockDim.x+threadIdx.x];
}
}