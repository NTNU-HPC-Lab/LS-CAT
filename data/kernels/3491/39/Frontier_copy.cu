#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Frontier_copy( unsigned int *frontier, unsigned int *frontier2, unsigned int *frontier_length)
{
unsigned int tid=threadIdx.x + blockDim.x * blockIdx.x;

if(tid<*frontier_length)
{
frontier[tid]=frontier2[tid];
}
if(tid==0)
{
g_mutex=0;
g_mutex2=0;
*g_q_offsets=0;
*g_q_size=0;
}
}