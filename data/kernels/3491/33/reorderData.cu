#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorderData(uint  startbit, uint  *outKeys, uint  *outValues, uint2 *keys, uint2 *values, uint  *blockOffsets, uint  *offsets, uint  *sizes, uint  totalBlocks)
{
uint GROUP_SIZE = blockDim.x;
__shared__ uint2 sKeys2[256];
__shared__ uint2 sValues2[256];
__shared__ uint  sOffsets[16];
__shared__ uint  sBlockOffsets[16];
uint* sKeys1   = (uint*) sKeys2;
uint* sValues1 = (uint*) sValues2;

uint blockId = blockIdx.x;

uint i = blockId * blockDim.x + threadIdx.x;

sKeys2[threadIdx.x]   = keys[i];
sValues2[threadIdx.x] = values[i];

if(threadIdx.x < 16)
{
sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks +
blockId];
sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
}
__syncthreads();

uint radix = (sKeys1[threadIdx.x] >> startbit) & 0xF;
uint globalOffset = sOffsets[radix] + threadIdx.x - sBlockOffsets[radix];

outKeys[globalOffset]   = sKeys1[threadIdx.x];
outValues[globalOffset] = sValues1[threadIdx.x];

radix = (sKeys1[threadIdx.x + GROUP_SIZE] >> startbit) & 0xF;
globalOffset = sOffsets[radix] + threadIdx.x + GROUP_SIZE -
sBlockOffsets[radix];

outKeys[globalOffset]   = sKeys1[threadIdx.x + GROUP_SIZE];
outValues[globalOffset] = sValues1[threadIdx.x + GROUP_SIZE];

}