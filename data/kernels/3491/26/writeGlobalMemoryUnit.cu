#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeGlobalMemoryUnit(float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
int s = gid*512;
for (j=0 ; j<repeat ; ++j)
{
output[(s+0)&(size-1)] = gid;
output[(s+1)&(size-1)] = gid;
output[(s+2)&(size-1)] = gid;
output[(s+3)&(size-1)] = gid;
output[(s+4)&(size-1)] = gid;
output[(s+5)&(size-1)] = gid;
output[(s+6)&(size-1)] = gid;
output[(s+7)&(size-1)] = gid;
output[(s+8)&(size-1)] = gid;
output[(s+9)&(size-1)] = gid;
output[(s+10)&(size-1)] = gid;
output[(s+11)&(size-1)] = gid;
output[(s+12)&(size-1)] = gid;
output[(s+13)&(size-1)] = gid;
output[(s+14)&(size-1)] = gid;
output[(s+15)&(size-1)] = gid;
s = (s+16)&(size-1);
}
}