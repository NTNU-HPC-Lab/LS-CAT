#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readLocalMemory(const float *data, float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
float sum = 0;
int tid=threadIdx.x, localSize=blockDim.x, grpid=blockIdx.x,
litems=2048/localSize, goffset=localSize*grpid+tid*litems;
int s = tid;
__shared__ float lbuf[2048];
for ( ; j<litems && j<(size-goffset) ; ++j)
lbuf[tid*litems+j] = data[goffset+j];
for (int i=0 ; j<litems ; ++j,++i)
lbuf[tid*litems+j] = data[i];
__syncthreads();
for (j=0 ; j<repeat ; ++j)
{
float a0 = lbuf[(s+0)&(2047)];
float a1 = lbuf[(s+1)&(2047)];
float a2 = lbuf[(s+2)&(2047)];
float a3 = lbuf[(s+3)&(2047)];
float a4 = lbuf[(s+4)&(2047)];
float a5 = lbuf[(s+5)&(2047)];
float a6 = lbuf[(s+6)&(2047)];
float a7 = lbuf[(s+7)&(2047)];
float a8 = lbuf[(s+8)&(2047)];
float a9 = lbuf[(s+9)&(2047)];
float a10 = lbuf[(s+10)&(2047)];
float a11 = lbuf[(s+11)&(2047)];
float a12 = lbuf[(s+12)&(2047)];
float a13 = lbuf[(s+13)&(2047)];
float a14 = lbuf[(s+14)&(2047)];
float a15 = lbuf[(s+15)&(2047)];
sum += a0+a1+a2+a3+a4+a5+a6+a7+a8+a9+a10+a11+a12+a13+a14+a15;
s = (s+16)&(2047);
}
output[gid] = sum;
}