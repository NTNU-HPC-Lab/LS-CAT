#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddUniform4(uint *d_vector, const uint *d_uniforms, const int n)
{
__shared__ uint uni[1];

if (threadIdx.x == 0)
{
uni[0] = d_uniforms[blockIdx.x];
}

unsigned int address = threadIdx.x + (blockIdx.x *
blockDim.x * 4);

__syncthreads();

// 4 elems per thread
for (int i = 0; i < 4 && address < n; i++)
{
d_vector[address] += uni[0];
address += blockDim.x;
}
}