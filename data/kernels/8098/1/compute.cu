#include "hip/hip_runtime.h"
#include "includes.h"


#define THREADS_PER_BLOCK 1024
#define TIME 3600000








__global__ void compute(float *a_d, float *b_d, float *c_d, int arraySize)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
float temp;
if( ix > 0 && ix < arraySize-1){
temp = (a_d[ix+1]+a_d[ix-1])/2.0;
__syncthreads();
b_d[ix]=temp;
__syncthreads();
}



}