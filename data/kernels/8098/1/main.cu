#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "compute.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *a_d = NULL;
hipMalloc(&a_d, XSIZE*YSIZE);
float *b_d = NULL;
hipMalloc(&b_d, XSIZE*YSIZE);
float *c_d = NULL;
hipMalloc(&c_d, XSIZE*YSIZE);
int arraySize = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
compute<<<gridBlock,threadBlock>>>(a_d,b_d,c_d,arraySize);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
compute<<<gridBlock,threadBlock>>>(a_d,b_d,c_d,arraySize);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
compute<<<gridBlock,threadBlock>>>(a_d,b_d,c_d,arraySize);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}