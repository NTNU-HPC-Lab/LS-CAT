#include "hip/hip_runtime.h"
#include "includes.h"


#define THREADS_PER_BLOCK 1024
#define TIME 3600000








__global__ void initialize(float *a_d, float *b_d, float *c_d, int arraySize)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
if(ix==0)
{
a_d[ix]=200.0;
b_d[ix]=200.0;

}

else if (ix<arraySize)
{
a_d[ix]=0.0;
b_d[ix]=0.0;
}

}