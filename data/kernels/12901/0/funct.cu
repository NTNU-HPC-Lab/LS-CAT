#include "hip/hip_runtime.h"
#include "includes.h"
/* Kintsakis Athanasios AEM 6667 */

#define inf 9999




__global__ void funct(int n, int k, float* x, int* qx)
{

int ix= blockIdx.x*blockDim.x + threadIdx.x;

//Epeksigisi
/*
float temp2=x[i*n+k] + x[k*n+j];
omws
i=ix/n;

kai
j=ix%n = ix&(n-1)
i*n = ix/n * n = ix-ix%n= ix-j

*/


int j=ix&(n-1);
float temp2=x[ix-j+k]+x[k*n+j];

if(x[ix]>temp2)
{
x[ix]=temp2;
qx[ix]=k;
}

if(x[ix]==inf)
{
qx[ix]=-2;
}

}