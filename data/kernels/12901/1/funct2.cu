#include "hip/hip_runtime.h"
#include "includes.h"
/* Kintsakis Athanasios AEM 6667 */

#define inf 9999




__global__ void funct2(int n, int k, float* x, int* qx)
{

int ix= blockIdx.x*blockDim.x + threadIdx.x;

int j=ix&(n-1);
float temp2=x[ix-j+k]+x[k*n+j];
if(x[ix]>temp2)
{
x[ix]=temp2;
qx[ix]=k;
}

}