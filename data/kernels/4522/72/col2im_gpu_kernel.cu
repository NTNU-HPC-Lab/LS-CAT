#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
int index = blockIdx.x*blockDim.x+threadIdx.x;
for(; index < n; index += blockDim.x*gridDim.x){
float val = 0;
int w = index % width + pad;
int h = (index / width) % height + pad;
int c = index / (width * height);
// compute the start and end of the output
int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
int w_col_end = min(w / stride + 1, width_col);
int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
int h_col_end = min(h / stride + 1, height_col);
// equivalent implementation
int offset =
(c * ksize * ksize + h * ksize + w) * height_col * width_col;
int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
int coeff_w_col = (1 - stride * height_col * width_col);
for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
}
}
data_im[index] += val;
}
}