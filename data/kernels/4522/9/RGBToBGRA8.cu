#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RGBToBGRA8(float3* srcImage, uchar4* dstImage, int width, int height, float scaling_factor)
{
const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

const int pixel = y * width + x;

if( x >= width )
return;

if( y >= height )
return;

const float3 px = srcImage[pixel];
dstImage[pixel] = make_uchar4(px.z * scaling_factor,
px.y * scaling_factor,
px.x * scaling_factor,
255.0f * scaling_factor);
}