#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RGBToRGBAf(uchar3* srcImage, float4* dstImage, int width, int height)
{
const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

const int pixel = y * width + x;

if( x >= width )
return;

if( y >= height )
return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);

const float  s  = 1.0f;
const uchar3 px = srcImage[pixel];

dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}