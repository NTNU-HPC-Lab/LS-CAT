#include "hip/hip_runtime.h"
#include "includes.h"


#define NX 100                          // No. of cells in x direction
#define NY 100                          // No. of cells in y direction
#define NZ 100                          // No. of cells in z direction
#define N (NX*NY*NZ)            // N = total number of cells in domain
#define L 100                             // L = length of domain (m)
#define H 100                             // H = Height of domain (m)
#define W 100                             // W = Width of domain (m)
#define DX (L/NX)                       // DX, DY, DZ = grid spacing in x,y,z.
#define DY (H/NY)
#define DZ (W/NZ)
#define DT 0.001                       // Time step (seconds)

#define R (1.0)           // Dimensionless specific gas constant
#define GAMA (7.0/5.0)    // Ratio of specific heats
#define CV (R/(GAMA-1.0)) // Cv
#define CP (CV + R)       // Cp

//#define DEBUG_VALUE

float *dens;              //density
float *temperature;        //temperature
float *xv;                //velocity in x
float *yv;                //velocity in y
float *zv;                //velocity in z
float *press;             //pressure

float *d_dens;              //density
float *d_temperature;       //temperature
float *d_xv;                //velocity in x
float *d_yv;                //velocity in y
float *d_zv;                //velocity in z
float *d_press;             //pressure

float *U;
float *U_new;
float *E;
float *F;
float *G;
float *FF;
float *FB;
float *FR;
float *FL;
float *FU;
float *FD;

float *h_body;
float *d_body;

int total_cells = 0;            // A counter for computed cells


__global__ void GPUHeatContactFunction(float *a, float *b, int *body) {

}