#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LeftRightBound2D(double *Hs, double *Ztopo, double *K2e, double *K2w, int BC2D, int M, int N) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < M) {
// no-flow BCs
if (BC2D == 0) {
Hs[tid*N] = Hs[tid*N+1];
Hs[(tid+1)*N-1] = Hs[(tid+1)*N-2];

} else {    // Critical depth flow BCs
Hs[tid*N] = hcri + Ztopo[tid*N];
Hs[(tid+1)*N-1] = hcri + Ztopo[(tid+1)*N-1];
}

K2w[tid*N] = K2w[tid*N+1];
K2e[(tid+1)*N-1] = K2e[(tid+1)*N-2];
tid += blockDim.x * gridDim.x;
}
}