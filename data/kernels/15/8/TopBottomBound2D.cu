#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TopBottomBound2D(double *Hs, double *Ztopo, double *K2n, double *K2s, int BC2D, int M, int N) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;

while (tid < N) {
// no-flow BCs
if (BC2D == 0) {
Hs[tid] = Hs[N+tid];
Hs[(M-1)*N+tid] = Hs[(M-2)*N+tid];

} else {    // Critical depth flow BCs
Hs[tid] = hcri + Ztopo[tid];
Hs[(M-1)*N+tid] = hcri + Ztopo[(M-1)*N+tid];
}

K2s[tid] = K2s[N+tid];
K2n[(M-1)*N+tid] = K2n[(M-2)*N+tid];

tid += blockDim.x * gridDim.x;
}
}