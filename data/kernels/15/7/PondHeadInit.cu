#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PondHeadInit(double *ph, int size) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < size) {
ph[tid] = psi_min;
tid += blockDim.x * gridDim.x;
}
}