#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SweHInit(double *var_in1, double *var_in2, double *var_out, int size) {
// Get thread id
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < size) {
// Transfer data and memory and calculation
var_out[tid] = var_in1[tid] - var_in2[tid];

// Thread id update
tid += blockDim.x * gridDim.x;
}
}