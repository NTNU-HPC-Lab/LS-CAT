#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyVariable(double *var_in, double *var_out, int size) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < size) {
// Transfer data and memory
var_out[tid] = var_in[tid];

// Update thread id if vector is long
tid += blockDim.x * gridDim.x;
}
}