#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void vanGenuchten(double *C, double *K, double *Ksat, double *theta, double *h, double n, double m, int size) {
double Se, h_, theta_;

for (int i = 0; i < size; i++) {
// Convert pressure unit from [m] to [cm]
h_ = h[i] * 100;

// Compute the volumetric moisture content [eqn 21]
if (h_ < 0) {
// for unsaturated soil conditions
theta_ = (theta_S - theta_R)/pow(1.0 + pow((-h_*alpha), n), m) + theta_R;
} else {
// for saturated soil condition h_ >= 0
theta_ = theta_S;
}
theta[i] = theta_;

// Compute the effective saturation [eqn 2]
Se = (theta_ - theta_R)/(theta_S - theta_R);

// Compute the hydraulic conductivity [eqn 8] - [Convert to unit: m/hr
K[i] = Ksat[i] * sqrt(Se) * (1.0 - pow(1.0-pow(Se, 1.0/m), m))
* (1.0 - pow(1.0-pow(Se, 1.0/m), m));

// Compute the specific moisture storage derivative of eqn (21).
// So we have to calculate C = d(theta)/dh. Then the unit is converted
// into [1/m].
if (h_ < 0) {
C[i] = 100 * -alpha * n * -1 * (1.0/n-1.0)*pow(alpha*abs(h_), n-1)
* (theta_R-theta_S) * pow(pow(alpha*abs(h_), n)+1, 1.0/n-2.0);
} else {
C[i] = 0.0;
}
}
}
__global__ void vanGenuchtenIntial(double *theta, double *K, double *Ksat, double *h, int size) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < size) {
double Se, _theta, _h, n, m, lambda;
n = nv;
lambda = n - 1.0;
m = lambda/n;

// Convert unit from [m] to [cm]
_h = h[i] * 100;

// Compute the volumetric moisture content [eqn 21]
if (_h < 0) {
_theta = (theta_S-theta_R) / pow(1.0+pow((alpha*(-_h)), n), m) + theta_R;
} else {
_theta = theta_S;
}
theta[i] = _theta;

// Compute the effective saturation [eqn 2]
Se = (_theta - theta_R)/(theta_S - theta_R);

// Compute the hydraulic conductivity [eqn 8] - Convert to unit: m/hr
K[i] = Ksat[i] * sqrt(Se) * (1.0 - pow(1.0-pow(Se, 1.0/m), m))
*(1.0 - pow(1.0 - pow(Se, 1.0/m), m));

// Update threads if vector is long
i += blockDim.x * gridDim.x;
}
}