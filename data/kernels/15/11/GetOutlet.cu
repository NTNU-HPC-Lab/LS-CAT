#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetOutlet(double *h, double *houtlet, double *u, double *uout, double *v, double *vout, int M, int N, int t) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int ind = 2;
while (tid < M) {
houtlet[t*M+tid] = h[(tid+1)*N-ind];
vout[t*M+tid] = v[(tid+1)*N-ind];
uout[t*M+tid] = u[(tid+1)*N-ind];
tid += blockDim.x * gridDim.x;
}
}