#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float sigmoid(float a) {
return 1.0 / (1.0 + exp (-a));
}
__global__ void sigmoid_kernel(float *vec, int len) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < len) {
vec[index] = sigmoid(vec[index]);
}
}