#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minus_one(float *matrix, unsigned int *indices, unsigned int row, unsigned int col) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < row)
matrix[index * col + indices[index]] -= 1;
}