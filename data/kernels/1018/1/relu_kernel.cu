#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float relu(float a) {
return a < 0 ? 0 : a;
}
__global__ void relu_kernel(float *vec, int len) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < len) {
vec[index] = relu(vec[index]);
}
}