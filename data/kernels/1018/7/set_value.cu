#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_value(float value, float *array, unsigned int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
array[index] = value;
}