#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float sigmoid(float a) {
return 1.0 / (1.0 + exp (-a));
}
__global__ void sigmoid_derivative(float *upper_grads, float *upper_values, unsigned int upper_size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < upper_size)
upper_grads[index] *= upper_values[index]*(1.0f - upper_values[index]);
}