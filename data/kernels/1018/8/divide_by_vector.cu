#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divide_by_vector(float *matrix, float *vector, unsigned int row, unsigned int col) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < row * col)
matrix[index] /= vector[index / col];
}