#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_values(float *num, size_t size, float abs_max)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
num[idx] = (abs_max + abs_max) * num[idx] - abs_max;
}