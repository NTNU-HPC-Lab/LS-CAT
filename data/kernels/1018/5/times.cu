#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times(float *input, unsigned int input_size, float *output, unsigned int n) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < n * input_size)
output[index] = input[index % input_size];
}