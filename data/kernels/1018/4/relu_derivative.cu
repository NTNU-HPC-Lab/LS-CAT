#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float relu(float a) {
return a < 0 ? 0 : a;
}
__global__ void relu_derivative(float *upper_grads, float *upper_values, unsigned int upper_size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < upper_size)
if (upper_values[index] == 0)
upper_grads[index] = 0;
}