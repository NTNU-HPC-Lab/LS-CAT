#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pick_minus_log_ps(float *matrix, float *minus_log_ps, unsigned int *indices, unsigned int row, unsigned int col) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < row)
minus_log_ps[index] = -log(matrix[index * col + indices[index]]);
}