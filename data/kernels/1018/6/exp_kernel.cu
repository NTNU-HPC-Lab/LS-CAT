#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_kernel(float *array, unsigned int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size)
array[index] = exp(array[index]);
}