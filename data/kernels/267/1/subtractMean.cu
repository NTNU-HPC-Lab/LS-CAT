#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtractMean(float* d_inputArray, uint64_t inputLength, float d_mean)
{
uint32_t globalThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;

if(globalThreadIndex >= inputLength)
return;

d_inputArray[globalThreadIndex] -= d_mean;
}