#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setDiagonalToZero(float* d_matrix, uint64_t columnsAndRows)
{
int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

//Check for out of bounds
if(absoluteThreadIdx >= columnsAndRows)
return;

//set diagonal element to zero
int matrixIndex = absoluteThreadIdx * columnsAndRows + absoluteThreadIdx;
d_matrix[matrixIndex] = 0;
}