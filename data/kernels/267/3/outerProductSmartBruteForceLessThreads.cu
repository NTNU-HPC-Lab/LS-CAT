#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void outerProductSmartBruteForceLessThreads(float* resultMatrix, float* vec, uint64_t vectorLength)
{
int col = (blockIdx.x * blockDim.x) + threadIdx.x; //column
int row = (blockIdx.y * blockDim.y) + threadIdx.y; //row

//check bounds
if(row >= vectorLength || col >= vectorLength)
return;

//transpose
if(row > col)
{
row = vectorLength - row;
col = row + col;
}

int index = (row * vectorLength + col) - (row * (row + 1)) / 2;

resultMatrix[index] = vec[row] * vec[col];
}