#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void outerProductSmartBruteForce(float* resultMatrix, float* vec, int vectorLength)
{
int col = (blockIdx.x * blockDim.x) + threadIdx.x; //column
int row = (blockIdx.y * blockDim.y) + threadIdx.y; //row


//check bounds
if(row >= vectorLength || col >= vectorLength || row > col)
return;

int index = (row * vectorLength + col) - (row * (row + 1)) / 2;

resultMatrix[index] += vec[row] * vec[col];

}