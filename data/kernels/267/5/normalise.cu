#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalise(float* result, unsigned int resultLength, float* amps, unsigned int* hits)
{
int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

if(absoluteThreadIdx > resultLength)
return;

result[absoluteThreadIdx] = amps[absoluteThreadIdx] / hits[absoluteThreadIdx / 4];
}