#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void parallelMeanUnroll2(float* d_inputArray, uint64_t inputLength, float* d_outputMean)
{
uint32_t localThreadIndex = threadIdx.x;
uint32_t sumDataIndex = blockIdx.x * blockDim.x * 2 + localThreadIndex; //The index of the piece of data that I will sum into this current block
uint32_t globalThreadIndex = blockDim.x * blockIdx.x + localThreadIndex;

//calculate a pointer to this threadBlocks data
float* localBlockPointer = d_inputArray + blockIdx.x * blockDim.x * 2;

//Add the next blockDim.x's worth of data into this block before we start reducing
//Bounds checking
if(sumDataIndex + blockDim.x < inputLength)
{
d_inputArray[sumDataIndex] += d_inputArray[sumDataIndex + blockDim.x];
}

//Wait for all threads on this block to complete
__syncthreads();

//Start reducing
//In-place, strided, reduction
for (uint32_t stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (localThreadIndex < stride)
{
localBlockPointer[localThreadIndex] += localBlockPointer[localThreadIndex + stride];
}
}

//Wait for all threads on this block to complete
__syncthreads();

//If this is the thread with the global index of one, calculate the mean
if(globalThreadIndex == 0)
{
//Clear the output just incase it isn't already
*d_outputMean = 0;

for(uint32_t i = 0; i < gridDim.x; ++i)
{
*d_outputMean += d_inputArray[ i * blockDim.x * 2]; //Times 2 because we take in 'two blocks' worth of data for each actual block
}

*d_outputMean =  *d_outputMean / (inputLength - 1);

//printf("Mean: %f\n", *d_outputMean);
}

}