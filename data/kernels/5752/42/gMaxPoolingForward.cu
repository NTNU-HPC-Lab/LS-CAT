#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if(tid >= outRows * outCols)
return;

int rowId = tid / outRows;
int colId = tid % outRows;

float* b = in + (rowId * inCols) + (colId * width);
float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

if(colId == outRows - 1) {
width = lastWidth;
}

float currentMax = b[0] * localMask[0];
for(int i = 1; i < width; ++i) {
if(b[i] * localMask[i] > currentMax) {
currentMax = b[i] * localMask[i];
}
}

out[rowId + (colId * outCols)] = currentMax;
}