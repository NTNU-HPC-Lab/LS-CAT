#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if(tid >= adjRows * adjCols)
return;

int rowId = tid / adjRows;
int colId = tid % adjRows;

float* b = in + (rowId * inCols) + (colId * width);

if(colId == adjRows - 1) {
width = lastWidth;
}

float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
size_t currentMaxIdx = 0;
for(int i = 1; i < width; ++i) {
if(b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
currentMaxIdx = i;
}
}

adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx]
+= adj[rowId + (colId * adjCols)];
}