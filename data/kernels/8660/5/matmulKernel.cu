#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmulKernel(float* mat1,float* mat2, float* matP,int dim)	{
int thread_x,thread_y,i;
thread_x=blockIdx.x*blockDim.x+threadIdx.x;
thread_y=blockIdx.y*blockDim.y+threadIdx.y;
if(thread_x<dim&&thread_y<dim)	{
float P_value=0.;
for(i=0;i<dim;i++)	{
P_value+=mat1[thread_y*dim+i]*mat2[i*dim+thread_x];
}
matP[thread_y*dim+thread_x]=P_value;
}
}