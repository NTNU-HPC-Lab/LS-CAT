#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adjacent_difference_simple(int *result, int *input)
{
// compute this thread's global index
unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

if(i > 0)
{
// each thread loads two elements from global memory
int x_i = input[i];
int x_i_minus_one = input[i-1];

// compute the difference using values stored in registers
result[i] = x_i - x_i_minus_one;
}
}