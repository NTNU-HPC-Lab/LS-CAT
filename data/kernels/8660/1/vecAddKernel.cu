#include "hip/hip_runtime.h"
#include "includes.h"
void vectorAdd(double* A, double* B,double* C,int n);

__global__ void vecAddKernel(double* A, double* B, double* C, int n)	{
int i=blockDim.x*blockIdx.x+threadIdx.x;
if(i<n) {
C[i]=A[i]+B[i];
}
}