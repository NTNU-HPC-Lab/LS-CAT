#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inclusive_scan(const unsigned int *input, unsigned int *result)
{
extern __shared__ unsigned int sdata[];

unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

// load input into __shared__ memory
unsigned int sum = input[i];
sdata[threadIdx.x] = sum;
__syncthreads();
for(int offset = 1; offset < blockDim.x; offset <<= 1)
{
if(threadIdx.x >= offset)
{
sum += sdata[threadIdx.x - offset];
}

// wait until every thread has updated its partial sum
__syncthreads();

// write my partial sum
sdata[threadIdx.x] = sum;

// wait until every thread has written its partial sum
__syncthreads();
}

// we're done! each thread writes out its result
result[i] = sdata[threadIdx.x];
}