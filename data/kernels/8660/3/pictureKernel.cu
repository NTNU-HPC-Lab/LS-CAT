#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pictureKernel(float* d_pix,int X, int Y) {
int thread_x=blockDim.x*blockIdx.x+threadIdx.x;
int thread_y=blockDim.y*blockIdx.y+threadIdx.y;
//	printf("thread_x=%d,blockDim.x=%d,blockIdx.x=%d,threadIdx=%d\n",thread_x,blockDim.x,blockIdx.x,threadIdx.x);
//	printf("thread_y=%d,blockDim.y=%d,blockIdx.y=%d,threadIdy=%d\n",thread_y,blockDim.y,blockIdx.y,threadIdx.y);
//	use this printf nvcc -arch compute_20 pixel.cu
if(thread_x<X&&thread_y<Y)	{
d_pix[thread_y*X+thread_x]*=2;
}
}