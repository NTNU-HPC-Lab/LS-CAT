#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float3 color(unsigned int depth, unsigned int maxDepth)
{
if(depth == maxDepth)
return make_float3(0.0f, 0.0f, 0.0f);
else
return make_float3(1.0f, 1.0f, 1.0f);
}
__device__ unsigned int mandelDepth(float cr, float ci, int maxDepth)
{
float zr = 0.0f;
float zi = 0.0f;
float zrSqr = 0.0f;
float ziSqr = 0.0f;

unsigned int i;

for (i = 0; i < maxDepth; i++)
{
zi = zr * zi;
zi += zi + ci;
zr = zrSqr - ziSqr + cr;
zrSqr = zr * zr;
ziSqr = zi * zi;

if (zrSqr + ziSqr > 4.0f) break;
}

return i;
}
__global__ void mandel(float* buffer, float xMin, float xMax, float yMin, float yMax, unsigned int maxDepth)
{
int nx = blockDim.x;
int ny = gridDim.x;
float dx = (xMax - xMin) / nx;
float dy = (yMax - yMin) / ny;
float x = xMin + (threadIdx.x + 0.5f) * dx;
float y = yMin + (blockIdx.x + 0.5f) * dy;

unsigned int depth = mandelDepth(x, y, maxDepth);
float3 depthColor = color(depth, maxDepth);

int index = blockIdx.x*blockDim.x + threadIdx.x;
buffer[3*index + 0] = depthColor.x;
buffer[3*index + 1] = depthColor.y;
buffer[3*index + 2] = depthColor.z;
}