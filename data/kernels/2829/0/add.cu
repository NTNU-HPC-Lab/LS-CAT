#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to add the elements of two arrays


__global__ void add(int n, float *x, float *y)
{
int index = threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < n; i += stride)
y[i] = x[i] + y[i];
}