#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
int ii = blockDim.x * blockIdx.x + threadIdx.x;
if (ii < n)
c[ii] = a[ii] + b[ii];
}