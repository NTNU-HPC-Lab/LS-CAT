#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int  GPUKernel_Position(int i,int j) {
if (i<j){
return j*(j+1)/2+i;
}
return i*(i+1)/2+j;
}
__global__ void GPUKernel_Vp(int a, int v,double * in,double * out) {

int blockid      = blockIdx.x*gridDim.y + blockIdx.y;
int id      = blockid*blockDim.x + threadIdx.x;

if ( id >= v*v*v ) return;

int  d = id%v;
int  b = (id-d)%(v*v)/v;
int  c = (id-d-b*v)/(v*v);

if ( b < a ) return;
if ( d > c ) return;

int cd   = GPUKernel_Position(c,d);
int vtri = v*(v+1)/2;

out[(b-a)*vtri+cd] = in[(b-a)*v*v+d*v+c] + in[(b-a)*v*v+c*v+d];
}