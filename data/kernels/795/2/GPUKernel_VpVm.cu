#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int  GPUKernel_Position(int i,int j) {
if (i<j){
return j*(j+1)/2+i;
}
return i*(i+1)/2+j;
}
__global__ void GPUKernel_VpVm(int a, int v,double * in,double * outp,double * outm) {

int blockid = blockIdx.x*gridDim.y + blockIdx.y;
int id      = blockid*blockDim.x + threadIdx.x;

int v2 = v*v;

if ( id >= v2*v ) return;

int  d = id%v;
int  b = (id-d)%(v2)/v;

if ( b < a ) return;

int bma = b - a;

int  c = (id-d-b*v)/(v2);

if ( d > c ) return;

int cd   = GPUKernel_Position(c,d);
int vtri = v*(v+1)/2;

outp[bma*vtri+cd] = in[bma*v2+d*v+c] + in[bma*v2+c*v+d];
outm[bma*vtri+cd] = in[bma*v2+d*v+c] - in[bma*v2+c*v+d];
}