#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPUKernel_VpVm_tiled(int a, int bstart, int bsize,int v,double * in,double * outp,double * outm) {

int blockid = blockIdx.x*gridDim.y + blockIdx.y;
int id      = blockid*blockDim.x + threadIdx.x;

int v2 = v*v;

if ( id >= v2*bsize ) return;

// id : b*v2+c*v+d

int  d = id%v;
int  c = (id-d)%(v*v)/v;

if ( d > c ) return;

//int  b = (id-d)%(v*bsize)/v;


//int  c = (id-d-b*v)/(bsize*v);
int  b = (id-d-c*v)/(v*v);

if ( b + bstart < a ) return;

int cd   = c*(c+1)/2 + d;
int vtri = v*(v+1)/2;
int bv2  = b*v2;

//outp[b*vtri+cd] = in[bv2+d*v+c] + in[bv2+c*v+d];
//outm[b*vtri+cd] = in[bv2+d*v+c] - in[bv2+c*v+d];
outp[b*vtri+cd] = in[bv2+d*v+c] + in[id];
outm[b*vtri+cd] = in[bv2+d*v+c] - in[id];
}