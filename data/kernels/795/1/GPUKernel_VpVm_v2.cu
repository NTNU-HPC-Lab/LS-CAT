#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int  GPUKernel_Position(int i,int j) {
if (i<j){
return j*(j+1)/2+i;
}
return i*(i+1)/2+j;
}
__global__ void GPUKernel_VpVm_v2(int a, int b,int v,double * in,double * outp,double * outm) {

int blockid = blockIdx.x*gridDim.y + blockIdx.y;
int id      = blockid*blockDim.x + threadIdx.x;

int v2 = v*v;

if ( id >= v2 ) return;

int  d = id%v;
int  c = (id-d)/v;

if ( d > c ) return;

int cd   = GPUKernel_Position(c,d);

outp[cd] = in[d*v+c] + in[c*v+d];
outm[cd] = in[d*v+c] - in[c*v+d];
}