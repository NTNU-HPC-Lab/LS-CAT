#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPUKernel_Iqdb(int a,int v,int nQ,double * in,double * out) {

int blockid = blockIdx.x*gridDim.y + blockIdx.y;
int id      = blockid*blockDim.x + threadIdx.x;

if ( id >= v*v*nQ ) return;

int  q = id%nQ;
int  d = (id-q)%(nQ*v)/nQ;
int  b = (id-q-d*nQ)/(nQ*v);

if ( b < a ) return;

int id2 = (b-a)*nQ*v+d*nQ+q;
out[id2] = in[id];

}