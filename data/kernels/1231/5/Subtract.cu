#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
const int x = blockIdx.x*SUBTRACT_W + threadIdx.x;
const int y = blockIdx.y*SUBTRACT_H + threadIdx.y;
int p = y*pitch + x;
if (x<width && y<height)
d_Result[p] = d_Data1[p] - d_Data2[p];
__syncthreads();
}