#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if((x < width) && (y < height))
{
float dpts[3];
float sims[3];

dpts[0] = odpt[y * odpt_p + x];
dpts[1] = idptLst[y * idptLst_p + x];
dpts[2] = idptAct[y * idptAct_p + x];

sims[0] = osim[y * odpt_p + x];
sims[1] = isimLst[y * isimLst_p + x];
sims[2] = isimAct[y * isimAct_p + x];

float outDpt = dpts[0];
float outSim = simThr;
outDpt = ((sims[1] < outSim) ? dpts[1] : outDpt);
outSim = ((sims[1] < outSim) ? sims[1] : outSim);
outDpt = ((sims[2] < outSim) ? dpts[2] : outDpt);
outSim = ((sims[2] < outSim) ? sims[2] : outSim);

// osim[y*osim_p+x]=(outSim<simThr?outSim:1.0f);
// odpt[y*odpt_p+x]=(outSim<simThr?outDpt:-1.0f);
osim[y * osim_p + x] = outSim;
odpt[y * odpt_p + x] = outDpt;
};
}