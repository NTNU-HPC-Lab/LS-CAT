#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y)
{
int x, y, z, w;
w = ii % d3;
ii = ii/d3;
z = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
w = w*scale_factor+off_x;
z = z*scale_factor+off_y;
d2 *= scale_factor;
d3 *= scale_factor;
return (((x*d1+y)*d2)+z)*d3+w;

}
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
int x, y, z, w;
w = ii % d3;
ii = ii/d3;
z = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
w = w/scale_factor;
z = z/scale_factor;
d2 /= scale_factor;
d3 /= scale_factor;
return (((x*d1+y)*d2)+z)*d3+w;

}
__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3)
{
// output offset:
long ii = threadIdx.x + blockDim.x * blockIdx.x;
ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
if (ii >= no_elements) return;
for (int i=0; i < scale_factor; i++){
for(int j=0; j < scale_factor; j++){
int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
gradInput_data[ii] += gradOutput_data[ipidx];
}
}
}