#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_psi_phi_Kernel(float* psi, float* phi, const float* gAbsIx, const float* gAbsIy, const float* gIx, const float* gIy, int nPixels, float norm_for_contrast_num, float norm_for_contrast_denom, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;


float psi_num = 0, psi_denom = 0;
float phi_num = 0, phi_denom = 0;
if (norm_for_contrast_num == 0)
{
psi_num = 1;
phi_num = 1;
}
else if (norm_for_contrast_num == 1)
{
psi_num = gAbsIx[x];
phi_num = gAbsIy[x];
}
else if (norm_for_contrast_num == 2)
{
psi_num = gAbsIx[x] * gAbsIx[x];
phi_num = gAbsIy[x] * gAbsIy[x];
}
else
{
psi_num = pow(gAbsIx[x], norm_for_contrast_num);
phi_num = pow(gAbsIy[x], norm_for_contrast_num);
}

if (norm_for_contrast_denom == 0)
{
psi_denom = 1;
phi_denom = 1;
}
else if (norm_for_contrast_denom == 1)
{
psi_denom = fabs(gIx[x]) + eps;
phi_denom = fabs(gIy[x]) + eps;
}
else if (norm_for_contrast_denom == 2)
{
psi_denom = gIx[x] * gIx[x] + eps;
phi_denom = gIy[x] * gIy[x] + eps;
}
else
{
psi_denom = pow(fabs(gIx[x]), norm_for_contrast_denom) + eps;
phi_denom = pow(fabs(gIy[x]), norm_for_contrast_denom) + eps;
}
psi[x] = psi_num / psi_denom;
phi[x] = phi_num / phi_denom;

}