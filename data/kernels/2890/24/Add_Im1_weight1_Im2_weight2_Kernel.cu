#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add_Im1_weight1_Im2_weight2_Kernel(float* output, const float* Im1, const float weight1, const float* Im2, const float weight2, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;

int offset = y*width + x;
for (int c = 0; c < nChannels; c++)
{
output[offset*nChannels + c] = Im1[offset*nChannels + c] * weight1 + Im2[offset*nChannels + c] * weight2;
}
}