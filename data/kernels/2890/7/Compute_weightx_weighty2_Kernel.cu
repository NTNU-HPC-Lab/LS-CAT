#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty2_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;
if (norm_for_smooth_term == 2)
{
weightx[x] = 1.0f;
weighty[x] = 1.0f;
}
else if (norm_for_smooth_term == 1)
{
weightx[x] = 1.0f / (absIx[x] + eps);
weighty[x] = 1.0f / (absIy[x] + eps);
}
else if (norm_for_smooth_term == 0)
{
weightx[x] = 1.0f / (absIx[x] * absIx[x] + eps);
weighty[x] = 1.0f / (absIy[x] * absIy[x] + eps);
}
else
{
weightx[x] = 1.0f / (pow(absIx[x], 2.0f - norm_for_smooth_term) + eps);
weighty[x] = 1.0f / (pow(absIy[x], 2.0f - norm_for_smooth_term) + eps);
}
}