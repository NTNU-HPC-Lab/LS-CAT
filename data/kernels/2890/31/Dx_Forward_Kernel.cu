#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Dx_Forward_Kernel(float* output, const float* input, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;

int offset = y*width + x;
if (x == width - 1)
{
for (int c = 0; c < nChannels; c++)
output[offset*nChannels + c] = 0;
}
else
{
for (int c = 0; c < nChannels; c++)
output[offset*nChannels + c] = input[(offset + 1)*nChannels + c] - input[offset*nChannels + c];
}
}