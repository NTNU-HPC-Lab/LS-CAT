#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Laplacian_Kernel(float* output, const float* input, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;

int offset = y*width + x;

for (int c = 0; c < nChannels; c++)
{
float value = 0;
if (x == 0)
{
value += input[(offset + 1)*nChannels + c] - input[offset*nChannels + c];
}
else if (x == width - 1)
{
value += input[(offset - 1)*nChannels + c] - input[offset*nChannels + c];
}
else
{
value += input[(offset + 1)*nChannels + c] + input[(offset - 1)*nChannels + c] - 2 * input[offset*nChannels + c];
}

if (y == 0)
{
value += input[(offset + width)*nChannels + c] - input[offset*nChannels + c];
}
else if (y == height - 1)
{
value += input[(offset - width)*nChannels + c] - input[offset*nChannels + c];
}
else
{
value += input[(offset + width)*nChannels + c] + input[(offset - width)*nChannels + c] - 2 * input[offset*nChannels + c];
}

output[offset*nChannels + c] = value;
}
}