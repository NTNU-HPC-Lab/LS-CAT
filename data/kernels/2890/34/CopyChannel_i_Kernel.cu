#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyChannel_i_Kernel(float* output, const float* input, const int i, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;
int offset = y*width + x;
output[offset] = input[offset*nChannels + i];
}