#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MedianFilterWithMask3x3_Kernel(float* output, const float* input, const int width, const int height, const int nChannels, const bool* keep_mask)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;
int offset = y*width + x;
if (keep_mask[offset])
{
for (int c = 0; c < nChannels; c++)
output[offset*nChannels + c] = input[offset*nChannels + c];
return;
}

float vals[9] = { 0 };
int count = 0;
for (int c = 0; c < nChannels; c++)
{
count = 0;
int start_x = ((x - 1) >= 0) ? (x - 1) : 0;
int end_x = ((x + 1) <= (width - 1)) ? (x + 1) : (width - 1);
int start_y = ((y - 1) >= 0) ? (y - 1) : 0;
int end_y = ((y + 1) <= (height - 1)) ? (y + 1) : (height - 1);
for (int ii = start_y; ii <= end_y; ii++)
{
for (int jj = start_x; jj <= end_x; jj++)
{
int cur_offset = ii*width + jj;
if (keep_mask[cur_offset])
{
vals[count++] = input[cur_offset*nChannels + c];
}
}
}
if (count == 0)
{
output[offset*nChannels + c] = 0;
}
else
{
int mid = (count + 1) / 2;
for (int pass = 0; pass < mid; pass++)
{
float max_val = vals[pass];
int max_id = pass;
for (int id = pass + 1; id < count; id++)
{
if (max_val < vals[id])
{
max_val = vals[id];
max_id = id;
}
}
vals[max_id] = vals[pass];
vals[pass] = max_val;
}
output[offset*nChannels + c] = vals[mid];
}
}
}