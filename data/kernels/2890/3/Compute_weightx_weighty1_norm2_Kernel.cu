#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty1_norm2_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;
weightx[x] = psi[x];
weighty[x] = phi[x];
}