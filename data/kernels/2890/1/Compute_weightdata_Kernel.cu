#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightdata_Kernel(float* weightdata, const float* I, const float* input, int nPixels, int nChannels, int c, float norm_for_data_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;

if (norm_for_data_term == 2)
{
weightdata[x] = 1;
}
else if (norm_for_data_term == 1)
{
weightdata[x] = 1.0f / (fabs(I[x] - input[x*nChannels + c]) + eps);
}
else
{
weightdata[x] = pow(fabs(I[x] - input[x*nChannels + c]) + eps, norm_for_data_term - 2);
}
}