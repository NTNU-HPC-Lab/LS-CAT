#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty1_normother_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;

weightx[x] = psi[x] / (pow(absIx[x], 2.0f - norm_for_smooth_term) + eps);
weighty[x] = phi[x] / (pow(absIy[x], 2.0f - norm_for_smooth_term) + eps);
}