#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Solve_redblack1_Kernel(float* output, const float* input, int width, int height, int nChannels, int c, const float* weightdata, const float* weightx, const float* weighty, float lambda, float omega, bool redflag)
{
int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int x = bx*blockDim.x + tx;
int y = by*blockDim.y + ty;
if (x >= width || y >= height)
return;

if ((y + x) % 2 == redflag)
return;

int offset = y*width + x;
int slice = width*nChannels;
int offset_c = offset*nChannels + c;
float coeff = 0, sigma = 0, weight = 0;
if (y > 0)
{
weight = lambda*weighty[offset - width];
coeff += weight;
sigma += weight * output[offset_c - slice];
}
if (y < height - 1)
{
weight = lambda*weighty[offset];
coeff += weight;
sigma += weight*output[offset_c + slice];
}
if (x > 0)
{
weight = lambda*weightx[offset - 1];
coeff += weight;
sigma += weight*output[offset_c - nChannels];
}
if (x < width - 1)
{
weight = lambda*weightx[offset];
coeff += weight;
sigma += weight*output[offset_c + nChannels];
}
coeff += weightdata[offset];
sigma += weightdata[offset] * input[offset_c];
if (coeff > 0)
output[offset_c] = sigma / coeff*omega + output[offset_c] * (1 - omega);
}