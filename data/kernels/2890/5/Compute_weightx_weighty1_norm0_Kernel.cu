#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty1_norm0_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;

weightx[x] = psi[x] / (absIx[x] * absIx[x] + eps);
weighty[x] = phi[x] / (absIy[x] * absIy[x] + eps);
}