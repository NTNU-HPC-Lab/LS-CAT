#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Addwith_Kernel(float* in_out_put, const float* other, const float weight, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;

int offset = y*width + x;
for (int c = 0; c < nChannels; c++)
{
in_out_put[offset*nChannels + c] += other[offset*nChannels + c] * weight;
}
}