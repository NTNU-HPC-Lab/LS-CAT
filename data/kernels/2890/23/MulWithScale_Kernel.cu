#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MulWithScale_Kernel(float* in_out_put, const float scale, const int width, const int height, const int nChannels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x >= width || y >= height)
return;

int offset = y*width + x;
for (int c = 0; c < nChannels; c++)
{
in_out_put[offset*nChannels + c] *= scale;
}
}