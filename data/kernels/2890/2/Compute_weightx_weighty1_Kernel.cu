#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty1_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;
if (norm_for_smooth_term == 2)
{
weightx[x] = psi[x];
weighty[x] = phi[x];
}
else if (norm_for_smooth_term == 1)
{
weightx[x] = psi[x] / (absIx[x] + eps);
weighty[x] = phi[x] / (absIy[x] + eps);
}
else if (norm_for_smooth_term == 0)
{
weightx[x] = psi[x] / (absIx[x] * absIx[x] + eps);
weighty[x] = phi[x] / (absIy[x] * absIy[x] + eps);
}
else
{
weightx[x] = psi[x] / (pow(absIx[x], 2.0f - norm_for_smooth_term) + eps);
weighty[x] = phi[x] / (pow(absIy[x], 2.0f - norm_for_smooth_term) + eps);
}
}