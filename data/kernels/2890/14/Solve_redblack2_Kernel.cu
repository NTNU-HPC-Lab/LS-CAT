#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Solve_redblack2_Kernel(float* output, const float* input, int width, int height, int nChannels, int c, const float* weightx, const float* weighty, float lambda, float omega, bool redflag)
{
int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int x = bx*blockDim.x + tx;
int y = by*blockDim.y + ty;
if (x >= width || y >= height)
return;

if ((y + x) % 2 == redflag)
return;

int offset = y*width + x;
int slice = width*nChannels;
int offset_c = offset*nChannels + c;
float coeff = 0, sigma = 0, weight = 0;
if (y > 0)
{
weight = lambda*weighty[offset - width];
coeff += weight;
sigma += weight * output[offset_c - slice];
}
if (y < height - 1)
{
weight = lambda*weighty[offset];
coeff += weight;
sigma += weight*output[offset_c + slice];
}
if (x > 0)
{
weight = lambda*weightx[offset - 1];
coeff += weight;
sigma += weight*output[offset_c - nChannels];
}
if (x < width - 1)
{
weight = lambda*weightx[offset];
coeff += weight;
sigma += weight*output[offset_c + nChannels];
}
coeff += 1;
sigma += input[offset_c];
if (coeff > 0)
output[offset_c] = sigma / coeff*omega + output[offset_c] * (1 - omega);
}