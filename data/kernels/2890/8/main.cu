#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Compute_weightx_weighty2_norm2_Kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *weightx = NULL;
hipMalloc(&weightx, XSIZE*YSIZE);
float *weighty = NULL;
hipMalloc(&weighty, XSIZE*YSIZE);
const float *absIx = NULL;
hipMalloc(&absIx, XSIZE*YSIZE);
const float *absIy = NULL;
hipMalloc(&absIy, XSIZE*YSIZE);
int nPixels = 1;
float norm_for_smooth_term =  XSIZE*YSIZE;
float eps = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Compute_weightx_weighty2_norm2_Kernel<<<gridBlock,threadBlock>>>(weightx,weighty,absIx,absIy,nPixels,norm_for_smooth_term,eps);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Compute_weightx_weighty2_norm2_Kernel<<<gridBlock,threadBlock>>>(weightx,weighty,absIx,absIy,nPixels,norm_for_smooth_term,eps);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Compute_weightx_weighty2_norm2_Kernel<<<gridBlock,threadBlock>>>(weightx,weighty,absIx,absIy,nPixels,norm_for_smooth_term,eps);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}