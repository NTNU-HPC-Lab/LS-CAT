#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty2_norm0_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int x = bx*blockDim.x + tx;
if (x >= nPixels)
return;

weightx[x] = 1.0f / (absIx[x] * absIx[x] + eps);
weighty[x] = 1.0f / (absIy[x] * absIy[x] + eps);
}