#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalized_aligned_dot_products(const double* A, const double divisor, const unsigned int m, const unsigned int n, double* QT)
{
int a = blockIdx.x * blockDim.x + threadIdx.x;
if (a < n) {
QT[a] = A[a + m - 1] / divisor;
}
}