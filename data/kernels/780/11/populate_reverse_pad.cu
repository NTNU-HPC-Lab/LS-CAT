#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void populate_reverse_pad(const double *Q, double *Q_reverse_pad, const double *mean, const int window_size, const int size)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
double mu = *mean;
if(tid < window_size) {
Q_reverse_pad[tid] = Q[window_size - 1 - tid] - mu;
}else if(tid < size){
Q_reverse_pad[tid] = 0;
}
}