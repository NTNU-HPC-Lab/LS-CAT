#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefix_sum_scan(uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size)
{
// Note: The first block is already correctly populated.
//       Start on the second block.
const uint element = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

if (element < array_size) {
const uint cluster_offset = dev_auxiliary_array[blockIdx.x + 1];
dev_main_array[element] += cluster_offset;
}
}