#include "hip/hip_runtime.h"
#include "includes.h"

//Limited version of checkCudaErrors from hip/hip_runtime_api.h

#define checkCudaErrors(val) check_errors( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int width, int length, hiprandState *rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if ((i >= width) || (j >= length)) {
return;
}
int index = j * width+ i;
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1984, index, 0, &rand_state[index]);
}