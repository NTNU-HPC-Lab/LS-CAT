#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
int i = threadIdx.y + blockDim.y * blockIdx.y;
int j = threadIdx.x + blockDim.x * blockIdx.x;

if (i >= h || j >= w) return;

const int pos = i * s + j;

float scale = normalization_factor[pos];

float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

image[pos] *= invScale;
}