#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kArgMaxRowwise(float* mat, float* target, unsigned int width, unsigned int height) {
__shared__ float max_vals[32];
__shared__ unsigned int max_args[32];
float cur_max = -2e38;
unsigned int cur_arg = 0;
float val = 0;

for (unsigned int i = threadIdx.x; i < width; i += 32) {
val = mat[blockIdx.x * width + i];

if (val > cur_max) {
cur_max = val;
cur_arg = i;
}
}

max_vals[threadIdx.x] = cur_max;
max_args[threadIdx.x] = cur_arg;

__syncthreads();

if (threadIdx.x == 0) {
cur_max = -2e38;
cur_arg = 0;

for (unsigned int i = 0; i < 32; i++)
if (max_vals[i] > cur_max) {
cur_max = max_vals[i];
cur_arg = max_args[i];
}

target[blockIdx.x] = cur_arg;
}
}