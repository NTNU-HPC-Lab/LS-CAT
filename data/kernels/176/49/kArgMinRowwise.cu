#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kArgMinRowwise(float* mat, float* target, unsigned int width, unsigned int height) {
__shared__ float min_vals[32];
__shared__ unsigned int min_args[32];
float cur_min = 2e38;
unsigned int cur_arg = 0;
float val = 0;

for (unsigned int i = threadIdx.x; i < width; i += 32) {
val = mat[blockIdx.x * width + i];

if (val < cur_min) {
cur_min = val;
cur_arg = i;
}
}

min_vals[threadIdx.x] = cur_min;
min_args[threadIdx.x] = cur_arg;

__syncthreads();

if (threadIdx.x == 0) {
cur_min = 2e38;
cur_arg = 0;

for (unsigned int i = 0; i < 32; i++)
if (min_vals[i] < cur_min) {
cur_min = min_vals[i];
cur_arg = min_args[i];
}

target[blockIdx.x] = cur_arg;
}
}