#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e, int ry_s, int ry_e, int rz_s, int rz_e, int x_step, int y_step, int z_step, int size_x, int size_y, int size_z, int buf_strides_x, int buf_strides_y, int buf_strides_z, int type_size, int dim, int OPS_soa) {

int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
(y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
(z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

if (OPS_soa) src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
else src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
dest += ((idx_z - rz_s) * z_step * buf_strides_z +
(idx_y - ry_s) * y_step * buf_strides_y +
(idx_x - rx_s) * x_step * buf_strides_x) *
type_size * dim;
for (int d = 0; d < dim; d++) {
memcpy(dest+d*type_size, src, type_size);
if (OPS_soa) src += size_x * size_y * size_z * type_size;
else src += type_size;
}
}
}