#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e, int ry_s, int ry_e, int rz_s, int rz_e, int x_step, int y_step, int z_step, int size_x, int size_y, int size_z, int buf_strides_x, int buf_strides_y, int buf_strides_z, int type_size, int dim, int OPS_soa) {

int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
(y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
(z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

if (OPS_soa) dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
else dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
src += ((idx_z - rz_s) * z_step * buf_strides_z +
(idx_y - ry_s) * y_step * buf_strides_y +
(idx_x - rx_s) * x_step * buf_strides_x) *
type_size * dim;
for (int d = 0; d < dim; d++) {
memcpy(dest, src + d*type_size, type_size);
if (OPS_soa) dest += size_x * size_y * size_z * type_size;
else dest += type_size;
}
}
}