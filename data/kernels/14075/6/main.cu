#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "copy_kernel_tobuf.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
char *dest = NULL;
hipMalloc(&dest, XSIZE*YSIZE);
char *src = NULL;
hipMalloc(&src, XSIZE*YSIZE);
int rx_s = 1;
int rx_e = 1;
int ry_s = 1;
int ry_e = 1;
int rz_s = 1;
int rz_e = 1;
int x_step = 1;
int y_step = 1;
int z_step = 1;
int size_x = XSIZE*YSIZE;
int size_y = XSIZE*YSIZE;
int size_z = XSIZE*YSIZE;
int buf_strides_x = 2;
int buf_strides_y = 2;
int buf_strides_z = 2;
int type_size = XSIZE*YSIZE;
int dim = 2;
int OPS_soa = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
copy_kernel_tobuf<<<gridBlock,threadBlock>>>(dest,src,rx_s,rx_e,ry_s,ry_e,rz_s,rz_e,x_step,y_step,z_step,size_x,size_y,size_z,buf_strides_x,buf_strides_y,buf_strides_z,type_size,dim,OPS_soa);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
copy_kernel_tobuf<<<gridBlock,threadBlock>>>(dest,src,rx_s,rx_e,ry_s,ry_e,rz_s,rz_e,x_step,y_step,z_step,size_x,size_y,size_z,buf_strides_x,buf_strides_y,buf_strides_z,type_size,dim,OPS_soa);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
copy_kernel_tobuf<<<gridBlock,threadBlock>>>(dest,src,rx_s,rx_e,ry_s,ry_e,rz_s,rz_e,x_step,y_step,z_step,size_x,size_y,size_z,buf_strides_x,buf_strides_y,buf_strides_z,type_size,dim,OPS_soa);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}