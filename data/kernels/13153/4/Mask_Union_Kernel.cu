#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Union_Kernel( int* A, int* B, int* devOut)
{
const int idx = blockDim.x*blockIdx.x + threadIdx.x;

if( A[idx] + B[idx] > 0)
devOut[idx] = 1;
else
devOut[idx] = 0;
}