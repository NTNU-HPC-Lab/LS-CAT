#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Invert_Kernel( int* A, int* devOut)
{
const int idx = blockDim.x*blockIdx.x + threadIdx.x;
devOut[idx] = 1 - A[idx];
}