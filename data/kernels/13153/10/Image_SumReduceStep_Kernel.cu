#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Image_SumReduceStep_Kernel( int* devBufIn, int* devBufOut, int  lastBlockSize)
{
// ONLY USE THIS FUNCTION WITH BLOCK SIZE = (256,1,1);
// NOTE: This method was originally written to use exactly the amt
//       of shared memory available for each block, but I believe
//       I later was told that cmd args use shared mem, which would
//       result in this method spilling over.  Need to check on that.
__shared__ char sharedMem[4096];
int* shmBuf1 = (int*)sharedMem;
int* shmBuf2 = (int*)&sharedMem[512];

int globalIdx = 512 * blockIdx.x + threadIdx.x;
int localIdx  = threadIdx.x;

shmBuf1[localIdx]     = 0;
shmBuf1[localIdx+256] = 0;
shmBuf2[localIdx]     = 0;
shmBuf2[localIdx+256] = 0;

if(blockIdx.x == gridDim.x-1)
{
if(localIdx+256 >= lastBlockSize) devBufIn[globalIdx+256] = 0;
if(localIdx     >= lastBlockSize) devBufIn[globalIdx]     = 0;
}

// Now we reduce each block of 512 values (256 threads) to a single number

shmBuf1[localIdx] = devBufIn[globalIdx] + devBufIn[globalIdx + 256]; __syncthreads();
if(localIdx < 128) shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+128]; __syncthreads();
if(localIdx < 64)  shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+64];  __syncthreads();
if(localIdx < 32)  shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+32];  __syncthreads();
if(localIdx < 16)  shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+16];  __syncthreads();
if(localIdx < 8)   shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+8];   __syncthreads();
if(localIdx < 4)   shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+4];   __syncthreads();
if(localIdx < 2)   shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+2];   __syncthreads();

// 2 -> 1
if(localIdx < 1)
devBufOut[blockIdx.x] = shmBuf2[localIdx] + shmBuf2[localIdx + 1];
__syncthreads();

}