#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Difference_Kernel( int* A, int* B, int* devOut)
{
const int idx = blockDim.x*blockIdx.x + threadIdx.x;

if(A[idx] == B[idx])
devOut[idx] = 0;
else
devOut[idx] = 1;

// Should test if the extra algebra ops are worth removing the if-statement
// Convert to {-1, +1}
//int aval = A[idx]*2 - 1;
//int bval = B[idx]*2 - 1;
//devOut[idx] = (aval*bval+1)/2;
}