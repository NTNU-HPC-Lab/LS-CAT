#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Intersect_Kernel( int* A, int* B, int* devOut)
{
const int idx = blockDim.x*blockIdx.x + threadIdx.x;
devOut[idx] = A[idx] * B[idx];
}