#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Sum_Kernel( int* A, int valCount, int* scalarOut)
{

const int localIdx    = threadIdx.x;
const int globalIdx   = blockDim.x*blockIdx.x + threadIdx.x;
const int blockIdxOut = blockIdx.x / blockDim.x;

while(valCount > 1)
{
int localCount = blockDim.x;
while(localCount > 1)
{
localCount = localCount / 2;
if(localIdx < localCount)
A[globalIdx] += A[globalIdx + localCount];
}

if(localIdx == 0)
A[blockIdxOut] = A[globalIdx];

valCount = valCount / blockDim.x;
}

if(globalIdx==0)
scalarOut[0] = A[0];
}