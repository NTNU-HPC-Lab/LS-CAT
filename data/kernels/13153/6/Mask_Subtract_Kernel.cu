#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  Mask_Subtract_Kernel( int* A, int* B, int* devOut)
{
const int idx = blockDim.x*blockIdx.x + threadIdx.x;
if( B[idx] == 0)
devOut[idx] = A[idx];
else
devOut[idx] = 0;
}