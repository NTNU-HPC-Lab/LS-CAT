#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 32

#define COMMENT "Centrist_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
unsigned char red, green, blue;
} PPMPixel;

typedef struct {
int x, y;
PPMPixel *data;
} PPMImage;

__global__ void mod_CENTRIST(PPMPixel *image_out, PPMPixel *image_cp, int columns, int rows, int *hist, int hist_len) {

int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

__shared__ int hist_private[512];
int hist_index = (threadIdx.y*TILE_WIDTH + threadIdx.x); //get index in shared histogram
if(hist_index < hist_len) hist_private[hist_index] = 0;
__syncthreads();
if(col < columns && row < rows)
{
//create and copy small chunks to shared memory
__shared__ unsigned char image_cp_private[TILE_WIDTH][TILE_WIDTH];

//convert to grayscale
int img_index = row * columns + col; //get index in original image
int grayscale = (image_cp[img_index].red*299 + image_cp[img_index].green*587 + image_cp[img_index].blue*114)/1000; //avoid float point errors

image_cp_private[threadIdx.y][threadIdx.x] = grayscale;

__syncthreads();
if(col < columns - 2 && row < rows - 2) //ignore first/last row/column
{
int r, c, rr, cc;
float mean = 0.0;
for(r = threadIdx.y, rr = row; r <= threadIdx.y + 2; r++, rr++)
for(c = threadIdx.x , cc = col; c <= threadIdx.x + 2; c++, cc++)
{
if(r < TILE_WIDTH && c < TILE_WIDTH)
{
mean += image_cp_private[r][c];
}
else
{
int grayscale_neigh = (image_cp[rr*columns + cc].red*299 + image_cp[rr*columns + cc].green*587 + image_cp[rr*columns + cc].blue*114)/1000;
mean += grayscale_neigh;
}
}
mean /= 9.0;
int value = 0, k = 8;
for(r = threadIdx.y, rr = row ; r <= threadIdx.y + 2; r++, rr++)
for(c = threadIdx.x, cc = col ; c <= threadIdx.x + 2; c++, cc++)
{
if(r < TILE_WIDTH && c < TILE_WIDTH)
{
if(1.0*image_cp_private[r][c] >= mean)
value |= 1<<k;
}
else
{
int grayscale_neigh = (image_cp[rr*columns + cc].red*299 + image_cp[rr*columns + cc].green*587 + image_cp[rr*columns + cc].blue*114)/1000;
if(grayscale_neigh >= mean)
value |= 1<<k;
}
k--;
}
int img_out_ind = row * (columns - 2) + col; //get index in ouput original
image_out[img_out_ind].red = image_out[img_out_ind].blue = image_out[img_out_ind].green = value;
atomicAdd(&(hist_private[value]), 1);
}
__syncthreads();
if(hist_index == 0)
{
for(int i = 0; i < hist_len; i++)
atomicAdd(&(hist[i]), hist_private[i]); //init shared histogram
}
}
}