#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ComputePositions.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *g_Data1 = NULL;
hipMalloc(&g_Data1, XSIZE*YSIZE);
float *g_Data2 = NULL;
hipMalloc(&g_Data2, XSIZE*YSIZE);
float *g_Data3 = NULL;
hipMalloc(&g_Data3, XSIZE*YSIZE);
int *d_Ptrs = NULL;
hipMalloc(&d_Ptrs, XSIZE*YSIZE);
float *d_Sift = NULL;
hipMalloc(&d_Sift, XSIZE*YSIZE);
int numPts = 1;
int maxPts = 1;
int w = XSIZE;
int h = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ComputePositions<<<gridBlock,threadBlock>>>(g_Data1,g_Data2,g_Data3,d_Ptrs,d_Sift,numPts,maxPts,w,h);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ComputePositions<<<gridBlock,threadBlock>>>(g_Data1,g_Data2,g_Data3,d_Ptrs,d_Sift,numPts,maxPts,w,h);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ComputePositions<<<gridBlock,threadBlock>>>(g_Data1,g_Data2,g_Data3,d_Ptrs,d_Sift,numPts,maxPts,w,h);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}