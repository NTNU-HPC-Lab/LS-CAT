#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
*
* Permission is hereby granted, free of charge, to any person obtaining a
* copy of this software and associated documentation files (the "Software"),
* to deal in the Software without restriction, including without limitation
* the rights to use, copy, modify, merge, publish, distribute, sublicense,
* and/or sell copies of the Software, and to permit persons to whom the
* Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
* THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
* FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;







}
__global__ void Subsample_Bilinear_ushort4(ushort4 *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
ushort4 c0 = tex2D(ushort4_tex, xi-dx, yi-dy);
ushort4 c1 = tex2D(ushort4_tex, xi+dx, yi-dy);
ushort4 c2 = tex2D(ushort4_tex, xi-dx, yi+dy);
ushort4 c3 = tex2D(ushort4_tex, xi+dx, yi+dy);
int4 res;
res.x =  ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
res.y =  ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
res.z =  ((int)c0.z+(int)c1.z+(int)c2.z+(int)c3.z+2) >> 2;
res.w =  ((int)c0.w+(int)c1.w+(int)c2.w+(int)c3.w+2) >> 2;
dst[yo*dst_pitch+xo] = make_ushort4(
(unsigned short)res.x, (unsigned short)res.y, (unsigned short)res.z, (unsigned short)res.w);
}
}