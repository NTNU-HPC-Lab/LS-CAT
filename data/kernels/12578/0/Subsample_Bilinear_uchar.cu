#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
*
* Permission is hereby granted, free of charge, to any person obtaining a
* copy of this software and associated documentation files (the "Software"),
* to deal in the Software without restriction, including without limitation
* the rights to use, copy, modify, merge, publish, distribute, sublicense,
* and/or sell copies of the Software, and to permit persons to whom the
* Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
* THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
* FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;







}
__global__ void Subsample_Bilinear_uchar(unsigned char *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
int y0 = tex2D(uchar_tex, xi-dx, yi-dy);
int y1 = tex2D(uchar_tex, xi+dx, yi-dy);
int y2 = tex2D(uchar_tex, xi-dx, yi+dy);
int y3 = tex2D(uchar_tex, xi+dx, yi+dy);
dst[yo*dst_pitch+xo] = (unsigned char)((y0+y1+y2+y3+2) >> 2);
}
}