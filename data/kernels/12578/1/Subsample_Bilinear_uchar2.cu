#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
*
* Permission is hereby granted, free of charge, to any person obtaining a
* copy of this software and associated documentation files (the "Software"),
* to deal in the Software without restriction, including without limitation
* the rights to use, copy, modify, merge, publish, distribute, sublicense,
* and/or sell copies of the Software, and to permit persons to whom the
* Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
* THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
* FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;







}
__global__ void Subsample_Bilinear_uchar2(uchar2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
uchar2 c0 = tex2D(uchar2_tex, xi-dx, yi-dy);
uchar2 c1 = tex2D(uchar2_tex, xi+dx, yi-dy);
uchar2 c2 = tex2D(uchar2_tex, xi-dx, yi+dy);
uchar2 c3 = tex2D(uchar2_tex, xi+dx, yi+dy);
int2 uv;
uv.x = ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
uv.y = ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
dst[yo*dst_pitch2+xo] = make_uchar2((unsigned char)uv.x, (unsigned char)uv.y);
}
}