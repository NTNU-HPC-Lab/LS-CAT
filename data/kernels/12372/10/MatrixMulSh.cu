#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{

//Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
__shared__ float Mds [TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds [TILE_WIDTH][TILE_WIDTH];

// calculate thread id
unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ ) // m indicate number of phase
{
Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)];
Nds[threadIdx.y][threadIdx.x] =  Nd[ ( m*TILE_WIDTH + threadIdx.y) * WIDTH + col];
__syncthreads() ; // for syncronizeing the threads

// Do for tile
for ( int k = 0; k<TILE_WIDTH ; k++ )
Pd[row*WIDTH + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y];
__syncthreads() ; // for syncronizeing the threads

}
}