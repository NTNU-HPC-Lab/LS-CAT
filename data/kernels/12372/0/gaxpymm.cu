#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p){
int bid = blockIdx.x;
int tid = threadIdx.x;
extern __shared__ double dots_s[];
if(bid<m)
if(tid<n){
for(int c=0;c<p;c++)
dots_s[bid*n*p+tid*p+c] = a[bid*n+tid] * *(b+(tid*p+c));
__syncthreads();
if(tid == 0){
for(int c=0;c<p;c++)
for(int i=1;i<n;i++){
dots_s[bid*n*p+c] +=dots_s[bid*n*p+i*p+c];
//			printf("y=%d, dots_s=%d, bid=%d, tid=%d, i=%d, n=%d\n",dots_s[bid*n], dots_s[bid*n+i],bid,tid,i,n);
}
for(int c=0;c<p;c++)
*(y+(bid*p+c))=dots_s[bid*n*p+c];
//		printf("y[%d]=%d, bid=%d, tid=%d\n",bid,y[bid],bid,tid);
}
}
}