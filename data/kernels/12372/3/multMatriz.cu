#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multMatriz(float *da, float *db, float *dc, int num){
float sum=0;
int j = threadIdx.x + blockIdx.x * blockDim.x;
int i = threadIdx.y + blockIdx.y * blockDim.y;
while(j<num){
while(i<num){
for (unsigned int k = 0; k<num; k++)
sum += da[i * num + k] * db[k * num + j];
dc[i*num + j] = (float) sum;
i += gridDim.y * blockDim.y;
}
j+=gridDim.x * blockDim.x;
i = threadIdx.y + blockIdx.y * blockDim.y;
}

}