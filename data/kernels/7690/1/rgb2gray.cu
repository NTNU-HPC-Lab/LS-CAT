#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2gray(float *grayImage, float *rgbImage, int channels, int width, int height) {
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x < width && y < height) {
// get 1D coordinate for the grayscale image
int grayOffset = y * width + x;
// one can think of the RGB image having
// CHANNEL times columns than the gray scale image
int rgbOffset = grayOffset * channels;
float r       = rgbImage[rgbOffset];     // red value for pixel
float g       = rgbImage[rgbOffset + 1]; // green value for pixel
float b       = rgbImage[rgbOffset + 2]; // blue value for pixel
// perform the rescaling and store it
// We multiply by floating point constants
grayImage[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
}
}