#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refreshClusters(dim3 *sum, dim3 *cluster, int *counter)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(counter[i] != 0) {
cluster[i].x = sum[i].x / counter[i];
cluster[i].y = sum[i].y / counter[i];
cluster[i].z = sum[i].z / counter[i];
} else {
cluster[i].z = cluster[i].x = cluster[i].z = 0;
}
sum[i] = dim3(0, 0, 0);
counter[i] = 0;
}