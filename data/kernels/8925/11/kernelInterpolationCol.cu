#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelInterpolationCol(double *result, int rows, int cols, int factor){
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

// Puntos de referencia para interpolacion
double a,b;
double   m;

//
// Interpolacion de columnas
// -------------------------
while (x < cols*factor && y<rows){
int trueY = y*factor;
int offset = x + trueY*cols*factor;

a = result[ offset                     ];
b = result[ offset + cols*factor*factor];

m = (b - a)/((double)factor);

// Antes de llegar al final
if (y != rows-1){
for(int p=0; p<=factor; ++p){
result[offset] = a;
a += m;
offset += cols*factor*factor;
}
}

// Borde final
else{
for(int p=0; p<factor; ++p){
result[offset] = b;
b -= m;
offset += cols*factor*factor;
}
}
}

}