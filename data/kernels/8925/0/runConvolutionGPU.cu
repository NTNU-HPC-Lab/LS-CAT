#include "hip/hip_runtime.h"
#include "includes.h"


// CUDA runtime

// Utilities and system includes

__constant__ double dev_kernel[KERNEL_LENGTH*KERNEL_LENGTH];

__global__ void runConvolutionGPU(double* image, double* result, int height, int width, int step)
{
int tx = threadIdx.x;
int ty = threadIdx.y;
//int O_TILE_WIDTH = blockDim.x-(lkernel/2)*2;
//int O_TILE_HEIGHT = blockDim.y-(lkernel/2)*2;
int row_o = threadIdx.y + blockIdx.y*O_TILE_HEIGHT;
int col_o = threadIdx.x + blockIdx.x*O_TILE_WIDTH;

int row_i = row_o - KERNEL_LENGTH/2;
int col_i = col_o - KERNEL_LENGTH/2;

__shared__ double N_ds[BLOCK_DIM_Y][BLOCK_DIM_X];

if((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < height)){
N_ds[ty][tx] = image[row_i*width+col_i];
}else{
N_ds[ty][tx] = 0.0f;
}

__syncthreads();

double output = 0.0f;
if(tx%step ==0 && ty%step==0 && ty < O_TILE_HEIGHT && tx < O_TILE_WIDTH){
for(int i=0; i<KERNEL_LENGTH; i++){
for(int j=0; j<KERNEL_LENGTH; j++){
output += dev_kernel[i*KERNEL_LENGTH+j]*N_ds[(i+ty)][(j+tx)];
}
}
if(row_o < height && col_o < width){
result[(row_o/step)*width/step+col_o/step] = output/9.0;
}
}
}