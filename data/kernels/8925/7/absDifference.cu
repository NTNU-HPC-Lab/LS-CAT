#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void absDifference(double *dDifference, double *dSup, double *dLow, int dSize){
int tid = threadIdx.x + blockIdx.x * blockDim.x;

while (tid < dSize) {
double a = dSup[tid];
double b = dLow[tid];
dDifference[tid] = (a > b) ? (a - b) : (b - a);
tid  += blockDim.x * gridDim.x;
}
}