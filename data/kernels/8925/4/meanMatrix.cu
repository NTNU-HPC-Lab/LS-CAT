#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void meanMatrix(double *dMatrix, double *dMean, int dSize, int *d_mutex){
__shared__ double cache[threadsPerBlock];
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int cacheIndex = threadIdx.x;
double temp = 0;
while (tid < dSize) {
temp += dMatrix[tid];
tid  += blockDim.x * gridDim.x;
}
// set the cache values
cache[cacheIndex] = temp;
// synchronize threads in this block
__syncthreads();

int i = blockDim.x/2;
while (i != 0) {
if (cacheIndex < i)
cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
i /= 2;
}

if(cacheIndex == 0){
while(atomicCAS(d_mutex,0,1) != 0);  //lock
*dMean += cache[0];
atomicExch(d_mutex, 0);  //unlock

*dMean = dMean[0]/dSize;
}
}