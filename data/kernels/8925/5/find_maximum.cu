#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_maximum(double *array, double *max, int dSize, int *d_mutex){
int index = threadIdx.x + blockIdx.x*blockDim.x;
int stride = gridDim.x*blockDim.x;
int offset = 0;

__shared__ double cache[threadsPerBlock];

double temp = -999999999.0;
while(index + offset < dSize){
temp = fmaxf(temp, array[index + offset]);
offset += stride;
}

cache[threadIdx.x] = temp;

__syncthreads();


// reduction
unsigned int i = blockDim.x/2;
while(i != 0){
if(threadIdx.x < i){
cache[threadIdx.x] = fmax(cache[threadIdx.x], cache[threadIdx.x + i]);
}

__syncthreads();
i /= 2;
}

if(threadIdx.x == 0){
while(atomicCAS(d_mutex,0,1) != 0);  //lock
*max = fmax(*max, cache[0]);
atomicExch(d_mutex, 0);  //unlock
}
}