#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum3(double *d_result, double *d_a, double *d_b, double *d_c, int dSize){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < dSize) {
d_result[tid] = d_a[tid] + d_b[tid] +d_c[tid];
tid  += blockDim.x * gridDim.x;
}
}