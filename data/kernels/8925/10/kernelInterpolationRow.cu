#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelInterpolationRow(double *original, double *result, int rows, int cols, int factor){
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int idOriginal,idResult;

// Puntos de referencia para interpolacion
double a,b;
double   m;

//
// Interpolacion de filas
// ----------------------
while (x < rows){
idOriginal = y*rows               + x       ;
idResult   = y*rows*factor*factor + x*factor;

a = original[ idOriginal    ];
b = original[ idOriginal + 1];

m = (b - a)/((double)factor);

// Antes de llegar al final
if (x != rows-1){
for(int p=0; p<=factor; ++p){
result[idResult] = a;
a += m;
++idResult;
}
}

// Borde final
else{
for(int p=0; p<factor; ++p){
result[idResult] = b;
b -= m;
++idResult;
}
}

}

}