#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divScalarMatrix(double *dMatrix, double *dScalar, int dSize){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < dSize) {
dMatrix[tid] = dMatrix[tid]/dScalar[0];
tid  += blockDim.x * gridDim.x;
}
}