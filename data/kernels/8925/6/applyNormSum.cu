#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void applyNormSum(double *dMap,double *dSupFeature, double *dMaxSupFeature, double *dMeanSupFeature, double *dInfFeature, double *dMaxInfFeature, double *dMeanInfFeature, int dSize){
int tid = threadIdx.x + blockIdx.x * blockDim.x;

double SupCoeff = (dMaxSupFeature[0] - dMeanSupFeature[0])*(dMaxSupFeature[0] - dMeanSupFeature[0]);
double InfCoeff = (dMaxInfFeature[0] - dMeanInfFeature[0])*(dMaxInfFeature[0] - dMeanInfFeature[0]);

while (tid < dSize) {
dMap[tid] += dSupFeature[tid]*SupCoeff + dInfFeature[tid]*InfCoeff;
tid  += blockDim.x * gridDim.x;
}
}