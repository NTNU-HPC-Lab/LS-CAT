#include "hip/hip_runtime.h"
#include "includes.h"
//header files included

//declaring the tile width and height
//for tile based matrix multiplication
#define TILE_WIDTH 32
#define TILE_HEIGHT 32

//Namespace for std
using namespace std;

//structure declaration for storing rows and columns for a matrix
struct matrix{
unsigned int rows;	//storing rows of a matrix
unsigned int cols;	//storing columns of a matrix
};

//handlerror declaration : to display file and line numbers of erroneous lines
__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3)
{
//shared memory takes one tile at a time
__shared__ float S1[TILE_WIDTH][TILE_HEIGHT];	//to store tiles for array 1
__shared__ float S2[TILE_HEIGHT][TILE_WIDTH];	//to store tiles for array 2

//threads x and y index for the current block
unsigned int tx=threadIdx.x;
unsigned int ty=threadIdx.y;

unsigned int c=blockIdx.x*blockDim.x + threadIdx.x;	//row value using x-index of current thread
unsigned int r=blockIdx.y*blockDim.y + threadIdx.y;	//column value using y-index of current thread

unsigned int idx=c*rows1+r;				//column major index, using row and column value

float val=0;		//register to store multiplication result initialized to zero

for(int m=0; m<1+((rows2-1)/TILE_WIDTH);m++)	//going over all tiles one by one, with each m
{

int var1=m*TILE_WIDTH+tx ;		//x thread value for current tile
int var2=m*TILE_WIDTH+ty ;		//y thread value for current tile

//copying a tile from array1
if (r < rows1 && var1 < rows2)		//if the value is associated to a valid matrix coordinate in array1 then store it to shared memory S1
S1[ty][tx]=array1[r + var1*rows1];//storing a "valid" value from array to shared memory
else
S1[ty][tx]=0;					//storing zero, since there is no valid value
__syncthreads();						//syncing all threads once shared memory S1 is stored

//copying a tile from array2
if(c < cols2 && var2 < rows2)	//if value is associates to a valid matrix coordinate in array2 then store it to shared memory S2
S2[ty][tx]=array2[var2+rows2*c];	//storing the valid value
else
S2[ty][tx]=0;		//storing zero, since no valid value
__syncthreads();		//synchronizing threads


for(int i=0; i<TILE_WIDTH;i++)	//going over entire tile, ty row in S1 and tx column in S2
val+=S1[ty][i]*S2[i][tx];	//and multiplying elements
__syncthreads();		//synchronizing threads

}

if(r < rows1 && c< cols2)	//removing degenerate cases
array3[idx]=val;	//saving multiplication result to global memory

}