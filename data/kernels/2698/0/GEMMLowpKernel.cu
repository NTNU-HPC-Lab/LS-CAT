#include "hip/hip_runtime.h"
#include "includes.h"




#define block_count 32
#define thread_per_block 1024
// Wrapper for ATen
__global__ void GEMMLowpKernel(const float* in, const int N, float* out, float scale, float shift, long long qmax, const float* noise, bool enforce_true_zero) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
out[i] = in[i];
if (enforce_true_zero)
out[i] = (out[i] / scale) + shift;
else
out[i] = (out[i] + shift) / scale;
out[i] += noise[i];
out[i] = fminf(out[i], qmax);
out[i] = fmaxf(out[i], 0.);
out[i] = roundf(out[i]);
if (enforce_true_zero)
out[i] = (out[i] - shift) * scale;
else
out[i] = out[i] * scale - shift;
}
}