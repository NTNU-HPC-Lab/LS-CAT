#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
int batch_index = blockIdx.x;
points += n*c*batch_index;
idx += m*nsample*batch_index;
out += m*nsample*c*batch_index;

int index = threadIdx.x;
int stride = blockDim.x;

for (int j=index;j<m;j+=stride) {
for (int k=0;k<nsample;++k) {
int ii = idx[j*nsample+k];
for (int l=0;l<c;++l) {
out[j*nsample*c+k*c+l] = points[ii*c+l];
}
}
}
}