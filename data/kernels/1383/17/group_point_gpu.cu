#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
int index = threadIdx.x;
points += n*c*index;
idx += m*nsample*index;
out += m*nsample*c*index;

for (int j=0;j<m;++j) {
for (int k=0;k<nsample;++k) {
int ii = idx[j*nsample+k];
for (int l=0;l<c;++l) {
out[j*nsample*c+k*c+l] = points[ii*c+l];
}
}
}
}