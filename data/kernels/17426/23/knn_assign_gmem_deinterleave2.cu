#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knn_assign_gmem_deinterleave2( uint32_t length, uint16_t k, uint32_t *neighbors) {
volatile uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
sample *= 2;
if (sample >= length) {
return;
}
for (int i = 0; i < k; i++) {
neighbors[sample * k + i] = neighbors[(length + sample) * k + k + i];
}
}