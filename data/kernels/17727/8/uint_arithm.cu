#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uint_arithm(float* A, float* B, float* C, unsigned int u1, unsigned int u2)
{
// device function call (warn if unsupported)
unsigned int _umin = umin ( u1, u2 );
// device function call (warn if unsupported)
unsigned int _umax = umax ( u1, u2 );
// device function call (warn if unsupported)
unsigned int _umin_global = ::umin ( u1, u2 );
// device function call (warn if unsupported)
unsigned int _umax_global = ::umax(u1, u2);
if (_umin != _umin_global) return;
if (_umax != _umax_global) return;
int i = threadIdx.x;
A[i] = i + _umin;
B[i] = i + _umax;
C[i] = A[i] + B[i];
}