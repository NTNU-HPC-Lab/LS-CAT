#include "hip/hip_runtime.h"
#include "includes.h"
/* CUDA API header files*/


extern "C"
__global__ void matrixMult(const double *Md, const double *Nd, double *Pd, int size)
{
int row = blockDim.x * blockIdx.x + threadIdx.x;
int col = blockDim.y * blockIdx.y + threadIdx.y;

if (row < size) {	// Don't do anything to the memory if we're above the size of the matrix
if (col < size) {

double Pvalue = 0;
for (int k = 0; k < size; k++) {
// Elements of 2d-arrays are stored in column-major ordering (i.e. column by column)
// This is a consequence of this code being called in R (where column-major ordering is the norm)
// whereas C usually stores 2d-array in row-major ordering
Pvalue += Md[k*size + row] * Nd[col*size + k];
}
Pd[col*size + row] = Pvalue;

}
}
}