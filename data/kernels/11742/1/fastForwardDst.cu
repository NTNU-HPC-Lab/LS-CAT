#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fastForwardDst(const int16_t* block, int16_t* coeff, int shift)  // input block, output coeff
{
int c[4];
int rnd_factor = 1 << (shift - 1);
int i = threadIdx.x;
// Intermediate Variables
c[0] = block[4 * i + 0] + block[4 * i + 3];
c[1] = block[4 * i + 1] + block[4 * i + 3];
c[2] = block[4 * i + 0] - block[4 * i + 1];
c[3] = 74 * block[4 * i + 2];

coeff[i] = (int16_t)((29 * c[0] + 55 * c[1] + c[3] + rnd_factor) >> shift);
coeff[4 + i] = (int16_t)((74 * (block[4 * i + 0] + block[4 * i + 1] - block[4 * i + 3]) + rnd_factor) >> shift);
coeff[8 + i] = (int16_t)((29 * c[2] + 55 * c[0] - c[3] + rnd_factor) >> shift);
coeff[12 + i] = (int16_t)((55 * c[2] - 29 * c[1] + c[3] + rnd_factor) >> shift);
}