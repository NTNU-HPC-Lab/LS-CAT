#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void negative_prob_multiply_csr_matrix_vector_kernel(unsigned int* cum_row_indexes, unsigned int* column_indexes, float* matrix_data, float* in_vector, float* out_vector, unsigned int outerdim) {

unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

if (row < outerdim) {
float prob = 1.0;

unsigned int row_start = cum_row_indexes[row];
unsigned int row_end = cum_row_indexes[row+1];

for (int i = row_start; i < row_end; i++) {
prob *= 1.0 - (matrix_data[i] * in_vector[column_indexes[i]]);
}
out_vector[row] = prob;
}
}