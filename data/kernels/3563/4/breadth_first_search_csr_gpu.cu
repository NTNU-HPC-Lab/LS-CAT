#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void breadth_first_search_csr_gpu(unsigned int* cum_row_indexes, unsigned int* column_indexes, int* matrix_data, unsigned int* in_infections, unsigned int* out_infections, unsigned int rows) {
unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

if (row < rows) {
if (in_infections[row] == 1) {
out_infections[row] = 1;

unsigned int row_start = cum_row_indexes[row];
unsigned int row_end = cum_row_indexes[row+1];

for (int i = row_start; i < row_end; i++) {
int timesteps_to_transmission = matrix_data[i];
if (timesteps_to_transmission != 0) {
if (timesteps_to_transmission == 1) {
out_infections[column_indexes[i]] = 1;
}
matrix_data[i] -= 1;
}
}
}
}
}