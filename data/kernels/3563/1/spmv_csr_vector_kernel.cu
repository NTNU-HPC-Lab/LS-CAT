#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void spmv_csr_vector_kernel(unsigned int computation_restriction_factor, const unsigned int* cum_row_indexes, const unsigned int* column_indexes, const float* matrix_data , const float* in_vector, float* out_vector, const unsigned int outerdim) {
__shared__ float vals[32];
int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
// global thread index
int warp_id = thread_id / 32;
// global warp index
int lane = thread_id & (32 - 1);
// thread index within the warp

int row = warp_id / computation_restriction_factor;
if (row < outerdim) {
int row_start = cum_row_indexes[row];
int row_end = cum_row_indexes[row+1];

// compute running prod per thread
vals[threadIdx.x] = 1;
for (int i = row_start + lane; i < row_end; i += 32) {
vals[threadIdx.x] *= 1 - (matrix_data[i] * in_vector[column_indexes[i]]);
}

// parallel reduction in shared memory
if (lane < 16) vals[threadIdx.x] *= vals[threadIdx.x + 16];
if (lane < 8) vals[threadIdx.x] *= vals[threadIdx.x + 8];
if (lane < 4) vals[threadIdx.x] *= vals[threadIdx.x + 4];
if (lane < 2) vals[threadIdx.x] *= vals[threadIdx.x + 2];
if (lane < 1) vals[threadIdx.x] *= vals[threadIdx.x + 1];

// first thread writes the result
if (lane == 0) out_vector[row] = vals[threadIdx.x];
}

}