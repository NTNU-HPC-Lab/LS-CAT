#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "graph_determ_weights.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned int *contact_mat_cum_row_indexes = NULL;
hipMalloc(&contact_mat_cum_row_indexes, XSIZE*YSIZE);
unsigned int *contact_mat_column_indexes = NULL;
hipMalloc(&contact_mat_column_indexes, XSIZE*YSIZE);
float *contact_mat_values = NULL;
hipMalloc(&contact_mat_values, XSIZE*YSIZE);
unsigned int rows = 1;
unsigned int values = 1;
float *immunities = NULL;
hipMalloc(&immunities, XSIZE*YSIZE);
float *shedding_curve = NULL;
hipMalloc(&shedding_curve, XSIZE*YSIZE);
unsigned int infection_length = 1;
float transmission_rate = 1;
int *infection_mat_values = NULL;
hipMalloc(&infection_mat_values, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
graph_determ_weights<<<gridBlock,threadBlock>>>(contact_mat_cum_row_indexes,contact_mat_column_indexes,contact_mat_values,rows,values,immunities,shedding_curve,infection_length,transmission_rate,infection_mat_values);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
graph_determ_weights<<<gridBlock,threadBlock>>>(contact_mat_cum_row_indexes,contact_mat_column_indexes,contact_mat_values,rows,values,immunities,shedding_curve,infection_length,transmission_rate,infection_mat_values);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
graph_determ_weights<<<gridBlock,threadBlock>>>(contact_mat_cum_row_indexes,contact_mat_column_indexes,contact_mat_values,rows,values,immunities,shedding_curve,infection_length,transmission_rate,infection_mat_values);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}