#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void graph_determ_weights(unsigned int* contact_mat_cum_row_indexes, unsigned int* contact_mat_column_indexes, float* contact_mat_values, unsigned int rows, unsigned int values, float* immunities, float* shedding_curve, unsigned int infection_length, float transmission_rate, int* infection_mat_values) {

unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

hiprandState state;
hiprand_init(1234 + row, 0, 0, &state);
if (row < rows) {
for (int j=contact_mat_cum_row_indexes[row]; j<contact_mat_cum_row_indexes[row+1]; j++) {
float pinf_noshed = contact_mat_values[j] * transmission_rate * (1.0 - immunities[contact_mat_column_indexes[j]]);
int delay;
for (delay=1; delay<infection_length+1; delay++) {
//hiprand_uniform(&state)
if (hiprand_uniform(&state) < pinf_noshed * shedding_curve[delay - 1]) {
break;
}
}
if (delay > infection_length) {
delay = -1;
}
infection_mat_values[j] = delay;
}
}
}