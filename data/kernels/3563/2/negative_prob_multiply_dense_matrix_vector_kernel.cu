#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void negative_prob_multiply_dense_matrix_vector_kernel(float* matrix, float* in_vector, float* out_vector, unsigned int outerdim, unsigned int innerdim) {
// We parallelize at the level of matrix rows,
unsigned int row = blockIdx.x*blockDim.x+threadIdx.x;

float prob = 1.0;

if (row < outerdim) {
// each thread computes one element of the output vector
for (int i = 0; i < innerdim; i++) {
prob *= 1.0 - (matrix[row * innerdim + i] * in_vector[i]);
}
out_vector[row] = prob;
}
}