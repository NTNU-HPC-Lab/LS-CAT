#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void finiteDiff(const int c, const double dt, const double dx, const int nt, const int nx, double *u, double *un) {

int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int t = 0; t < nt; t++) {

for (int i = index; i < nx; i += stride) {
un[i] = u[i];
}

for (int i = index + 1; i < nx; i += stride) {
u[i] = un[i] - c * dt / dx * (un[i] - un[i - 1]);
}
}
}