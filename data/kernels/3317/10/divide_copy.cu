#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divide_copy(double *dest, const double *src, int length, const double divisor)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
double factor = 1.0 / divisor;
while (tid < length) {
dest[tid] = src[tid] * factor;
tid += blockDim.x * gridDim.x;
}
}