#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void diff_reduce(double *dev_w, double *feat, double *pos, int feat_dim, int pos_dim, int par0, int par1, int n_patch)
{
int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

double feat_dist = 0.0; // running entry sum of d_ij
double pos_dist = 0.0;  // running entry sum of f_ij
int feat_offi = i * feat_dim; // offset of x_i
int feat_offj = j * feat_dim; // offset of x_j
int pos_offi = i * pos_dim;   // offset of p_i
int pos_offj = j * pos_dim;   // offset of p_j
double feat_i, feat_j, pos_i, pos_j;
// temporary local variables for entry sum calculation
int k;

if (i == j || i >= n_patch || j >= n_patch)
return;

/* thread (i, j) computes W_ij */

// get the k-th element of difference vector d_ij
// and add it to feat_dist
for (k = 0; k < feat_dim; k++) {
feat_i = feat[feat_offi + k];
feat_j = feat[feat_offj + k];
feat_dist += (feat_i - feat_j) * (feat_i - feat_j);
}

// get the k-th element of difference vector f_ij
// and add it to pos_dist
for (k = 0; k < pos_dim; k++) {
pos_i = pos[pos_offi + k];
pos_j = pos[pos_offj + k];
pos_dist += (pos_i - pos_j) * (pos_i - pos_j);
}

dev_w[i + j * n_patch]
= exp( -feat_dist / (feat_dim * par0 * par0))
* exp( -pos_dist / (pos_dim * par1 * par1));
}