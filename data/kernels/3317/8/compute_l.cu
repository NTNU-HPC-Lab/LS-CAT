#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_l(double *dev_w, int n_patch)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int N = n_patch * n_patch;
while (tid < N) {
dev_w[tid] = ((tid % (n_patch + 1) == 0) ? 1.0 : 0.0) - dev_w[tid];
tid += blockDim.x * gridDim.x;
}
}