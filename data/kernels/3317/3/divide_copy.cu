#include "hip/hip_runtime.h"
#include "includes.h"

/*
* lanczos computes the smallest n_eigs eigenvalues for dev_L and the
* corresponding eigenvectors using the Lanczos algorithm.
*
* F: an array (n_patch by n_eigs) to store the eigenvectors
* Es: an array (1 by n_eigs) to store the eigenvalues
* dev_L: an array (n_patch by n_patch) representing the Laplacian matrix
* n_patch: the dimension of dev_L
*/
static double norm2(double *v, int length);

__global__ void divide_copy(double *dest, const double *src, int length, const double divisor)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
double factor = 1.0 / divisor;
while (tid < length) {
dest[tid] = src[tid] * factor;
tid += blockDim.x * gridDim.x;
}
}