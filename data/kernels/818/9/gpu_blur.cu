#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_blur(unsigned char* Pout, unsigned char* Pin, int width, int height){
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;
int k_size = 3;

if (col < width && row < height){
int pixVal = 0;
int pixels = 0;

for(int blurRow = -k_size; blurRow < k_size+1; blurRow++){
for(int blurCol = -k_size; blurCol < k_size+1; blurCol++){
int curRow = row + blurRow;
int curCol = col + blurCol;

if (curRow > -1 && curRow < height && curCol > -1 && curCol < width){
pixVal += Pin[curRow * width + curCol];
pixels++;
}
}
}

Pout[row * width + col] = (unsigned char) (pixVal / pixels);
}
}