#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_grey_and_thresh(unsigned char* Pout, unsigned char* Pin, int width, int height){

int channels = 3;
unsigned char thresh = 157;
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

// check if pixel within range
if (col < width && row < height){
int gOffset = row * width + col;
int rgbOffset = gOffset * channels;
unsigned char r = Pin[rgbOffset  ];
unsigned char g = Pin[rgbOffset+1];
unsigned char b = Pin[rgbOffset+2];
unsigned char gval = 0.21f*r + 0.71f*g + 0.07f*b;

if(gval > thresh){
Pout[gOffset] = 255;
}
else {
Pout[gOffset] = 0;
}
}
}