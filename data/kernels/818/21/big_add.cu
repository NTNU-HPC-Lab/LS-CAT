#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void big_add(int *a, int *b, int *c, unsigned int N){
// init thread id
int tid;
tid = blockIdx.x * blockDim.x + threadIdx.x;
// stride is for big arrays, i.e. bigger than threads we have
int stride = blockDim.x * gridDim.x;

// do the operations
while(tid < N){
c[tid] = a[tid] + b[tid];
tid += stride;
}
}