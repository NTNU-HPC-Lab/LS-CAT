#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void big_add(int *a, int *b, int *c, unsigned int N){
int tid;
tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
while(tid < N){
c[tid] = a[tid] + b[tid];
tid += stride;
}
}