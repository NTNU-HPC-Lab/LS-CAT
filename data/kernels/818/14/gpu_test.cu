#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_test(unsigned char* Pout, unsigned char* Pin, int width, int height) {
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;
int i = row * width + col;

if (row < height && col < width) {
Pout[i] = Pin[i];
}

}