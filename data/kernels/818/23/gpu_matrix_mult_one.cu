#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult_one(int *a, int *b, int *c, int m, int n, int k)
{
int row = blockIdx.y * blockDim.y + threadIdx.y; // get the row
int col = blockIdx.x * blockDim.x + threadIdx.x; // get the column
int sum = 0; // initialize the sum

if( col < k && row < m) // check to make sure that the thread needs to compute
{
for(int i = 0; i < n; i++)
{
sum += a[row * n + i] * b[i * k + col];
}
c[row * k + col] = sum;
}
}