#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult_two(int *d_M, int *d_N, int *d_P, int m, int n, int k)
{

// shared memory for tiling
__shared__ int Mds [TILE_WIDTH][TILE_WIDTH];
__shared__ int Nds [TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

// recall that TILE_WIDTH = blockDim
int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

int pval = 0;

// this loop is iterating through cols of M and rows of N
// recall that n is the shared inner dimension, that's why we're using it
// to define our loop size
for (int ph = 0; ph < n / TILE_WIDTH; ph++){
// boundary check for shared Mds
if (row < k && ph * TILE_WIDTH + tx < m){
// saving tile from M
/* indexing thought exercise:
* "row * k" gets us to our desired row in M
* adding "ph * TILE_WIDTH" moves our tile over to the desired tile location
* adding "tx" moves us to the desired location within the tile
* */
Mds[ty][tx] = d_M[row * k + ph * TILE_WIDTH + tx];
}
// boundary check
if (ph*TILE_WIDTH + ty < k && col < m){
// saving tile from N
/* indexing thought exercise:
* "ph * TILE_WIDTH" moves the tile "down" to the desired location
* adding "ty" gets us to the desired location within the tile
* multiplying by "k" does the magic (remember row major order)
* adding col moves the tile to the desired column*/
Nds[ty][tx] = d_N[(ph * TILE_WIDTH + ty) * k + col];
}

__syncthreads();  // execution barrier

for (int j = 0; j < TILE_WIDTH; j++){
// performing part of inner product
pval += Mds[ty][j] * Nds[j][tx];
}

__syncthreads();
}
if (row < k && col < m){
d_P[row * k + col] = pval;
}
}