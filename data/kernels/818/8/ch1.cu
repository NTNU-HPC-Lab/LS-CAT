#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ch1(unsigned char* Pout, unsigned char* Pin, int width, int height) {

int channels = 3;
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

// check if pixel within range
if (col < width && row < height){
int gOffset = row * width + col;
int rgbOffset = gOffset * channels;
unsigned char r = Pin[rgbOffset  ];
unsigned char g = Pin[rgbOffset+1];
unsigned char b = Pin[rgbOffset+2];
Pout[gOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
}
}