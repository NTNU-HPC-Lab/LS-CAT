#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *a, int *res){
// create shared memory for the threads in the block
__shared__ int cache[threadsPerBlock];

// get the thread id
int tid = threadIdx.x + blockIdx.x * blockDim.x;

// index into the cache for this block
int cacheIndex = threadIdx.x;

// set the value in cache
cache[cacheIndex] = a[tid];

__syncthreads(); //synchronize threads before continuing

int i = blockDim.x/2; // only want first half to do work
while( i != 0 ){
if (cacheIndex < i) // make sure we are not doing bogus add

// add the current index and ith element
cache[cacheIndex] += cache[cacheIndex + i];

__syncthreads(); // we want all threads to finish
i /= 2;
}
if (cacheIndex == 0) // only one thread needs to do this
*res = cache[0];
}