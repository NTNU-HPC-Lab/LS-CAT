#include "hip/hip_runtime.h"
#include "includes.h"

const float REAL_VALUE_MAX = 1000000.0f;
const int NUM_THREADS = 32;
const int SIZE = 10000;
const int DIMENSION = 2;

__device__ float clamp(float v, float mn = -REAL_VALUE_MAX, float mx = REAL_VALUE_MAX) {
return v < mn ? mn : v > mx ? mx : v;
}
__global__ void updateParticleKernel(float* P, float* V, float* PB, float* GB, float momentum, float introvert, float extrovert, float clamp_min, float clamp_max) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < SIZE * DIMENSION) {
P[i] = clamp(P[i] + V[i], clamp_min, clamp_max);
V[i] = clamp(momentum * V[i] + introvert * (PB[i] - P[i]) + extrovert * (GB[i % DIMENSION] - P[i]), clamp_min, clamp_max);
}
}