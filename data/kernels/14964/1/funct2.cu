#include "hip/hip_runtime.h"
#include "includes.h"
/**
#Copyright 2013 Athanassios Kintsakis

#Licensed under the Apache License, Version 2.0 (the "License");
#you may not use this file except in compliance with the License.
#You may obtain a copy of the License at

#http://www.apache.org/licenses/LICENSE-2.0

#Unless required by applicable law or agreed to in writing, software
#distributed under the License is distributed on an "AS IS" BASIS,
#WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
#See the License for the specific language governing permissions and
#limitations under the License.

# Author: Athanassios Kintsakis
# contact: akintsakis@issel.ee.auth.gr, athanassios.kintsakis@gmail.com
**/
#define inf 9999





__global__ void funct2(int n, int k, float* x, int* qx) {
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int j = ix & (n - 1);
float temp2 = x[ix - j + k] + x[k * n + j];
if (x[ix] > temp2) {
x[ix] = temp2;
qx[ix] = k;
}
}