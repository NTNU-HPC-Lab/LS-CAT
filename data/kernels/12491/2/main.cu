#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "rgb_to_xyY.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_r = NULL;
hipMalloc(&d_r, XSIZE*YSIZE);
float *d_g = NULL;
hipMalloc(&d_g, XSIZE*YSIZE);
float *d_b = NULL;
hipMalloc(&d_b, XSIZE*YSIZE);
float *d_x = NULL;
hipMalloc(&d_x, XSIZE*YSIZE);
float *d_y = NULL;
hipMalloc(&d_y, XSIZE*YSIZE);
float *d_log_Y = NULL;
hipMalloc(&d_log_Y, XSIZE*YSIZE);
float delta = 1;
int num_pixels_y = 1;
int num_pixels_x = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
rgb_to_xyY<<<gridBlock,threadBlock>>>(d_r,d_g,d_b,d_x,d_y,d_log_Y,delta,num_pixels_y,num_pixels_x);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
rgb_to_xyY<<<gridBlock,threadBlock>>>(d_r,d_g,d_b,d_x,d_y,d_log_Y,delta,num_pixels_y,num_pixels_x);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
rgb_to_xyY<<<gridBlock,threadBlock>>>(d_r,d_g,d_b,d_x,d_y,d_log_Y,delta,num_pixels_y,num_pixels_x);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}