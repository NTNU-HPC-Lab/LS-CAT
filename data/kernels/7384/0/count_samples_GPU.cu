#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define nsamples 250000
#define threadsPerBlock 500
#define num_blocks 500

// function to count samples in circle using cpu
__global__ void count_samples_GPU(float *d_X, float *d_Y, int *d_countInBlocks, int num_block, int samples)
{
__shared__ int shared_blocks[500];            // shared memory for threads in the same block

int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * num_block;

int inCircle = 0;
for (int i = index; i < samples; i += stride) {
float xValue = d_X[i];
float yValue = d_Y[i];

if (xValue*xValue + yValue * yValue <= 1.0f) {
inCircle++;
}
}

shared_blocks[threadIdx.x] = inCircle;
__syncthreads();                               //  prevent RAW/WAR/WAW hazards

// Pick thread 0 for each block to collect all points from each Thread.
if (threadIdx.x == 0)
{
int totalInCircleForABlock = 0;
for (int j = 0; j < blockDim.x; j++)
{
totalInCircleForABlock += shared_blocks[j];
}
d_countInBlocks[blockIdx.x] = totalInCircleForABlock;
}
}