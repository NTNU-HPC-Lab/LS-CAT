#include "hip/hip_runtime.h"
#include "includes.h"
/* This code will generate a fractal image. Uses OpenCV, to compile:
nvcc CudaFinal.cu `pkg-config --cflags --libs opencv`  */


typedef enum color {BLUE, GREEN, RED} Color;




__global__ void convert_to_rgb(float *hsv, unsigned char *dest, int width, int heigth, int step, int channels) {
float r, g, b;
float h, s, v;
int ren,col;

ren = blockIdx.x;
col = threadIdx.x;
h = hsv[(ren * step) + (col * channels) + RED];
s = hsv[(ren * step) + (col * channels) + GREEN];
v = hsv[(ren * step) + (col * channels) + BLUE];

float f = h/60.0f;
float hi = floorf(f);
f = f - hi;
float p = v * (1 - s);
float q = v * (1 - s * f);
float t = v * (1 - s * (1 - f));

if(hi == 0.0f || hi == 6.0f) {
r = v;
g = t;
b = p;
} else if(hi == 1.0f) {
r = q;
g = v;
b = p;
} else if(hi == 2.0f) {
r = p;
g = v;
b = t;
} else if(hi == 3.0f) {
r = p;
g = q;
b = v;
} else if(hi == 4.0f) {
r = t;
g = p;
b = v;
} else {
r = v;
g = p;
b = q;
}

dest[(ren * step) + (col * channels) + RED] =  (unsigned char) __float2uint_rn(255.0f * r);
dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) __float2uint_rn(255.0f * g);
dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) __float2uint_rn(255.0f * b);
}