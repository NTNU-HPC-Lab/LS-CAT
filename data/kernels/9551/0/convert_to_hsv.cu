#include "hip/hip_runtime.h"
#include "includes.h"
/* This code will generate a fractal image. Uses OpenCV, to compile:
nvcc CudaFinal.cu `pkg-config --cflags --libs opencv`  */


typedef enum color {BLUE, GREEN, RED} Color;




__global__ void convert_to_hsv(unsigned char *src, float *hsv, int width, int heigth, int step, int channels) {
float r, g, b;
float h, s, v;
int ren,col;

ren = blockIdx.x;
col = threadIdx.x;

r = src[(ren * step) + (col * channels) + RED] / 255.0f;
g = src[(ren * step) + (col * channels) + GREEN] / 255.0f;
b = src[(ren * step) + (col * channels) + BLUE] / 255.0f;

float max = fmax(r, fmax(g, b));
float min = fmin(r, fmin(g, b));
float diff = max - min;

v = max;

if(v == 0.0f) { // black
h = s = 0.0f;
} else {
s = diff / v;
if(diff < 0.001f) { // grey
h = 0.0f;
} else { // color
if(max == r) {
h = 60.0f * (g - b)/diff;
if(h < 0.0f) { h += 360.0f; }
} else if(max == g) {
h = 60.0f * (2 + (b - r)/diff);
} else {
h = 60.0f * (4 + (r - g)/diff);
}
}
}
// confusion line
float minh=40.0f;
float maxh=200.0f;
float minis = 0;
float maxs = 100;
float miniv = 0;
float maxv = 100;

// if conditionals to check the color blindness line, if the pixel is in this line i change the color to other color base shifting the h
if (h > minh && h < maxh && s > minis && s < maxs && v > miniv && v < maxv){

hsv[(ren * step) + (col * channels) + RED] =  (float) (h + 140.0f);
hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
} else { // this keep the pixel if it is out of the color blindnessline
hsv[(ren * step) + (col * channels) + RED] =  (float) (h);
hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
}


}