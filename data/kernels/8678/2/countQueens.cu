#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void countQueens(int* frontQueensPos, int* data, int* numFQP)
{
int localResult = 0;
//printf("%d\n", numFQP[0]);
int thisThread = ((blockIdx.x * gridDim.x + blockIdx.y) * gridDim.y + threadIdx.x)* blockDim.x + threadIdx.y;
//	printf("1_%d %d %d %d %d %d %d %d\n", thisThread, blockIdx.x, gridDim.x, blockIdx.y, gridDim.y, threadIdx.x, blockDim.x, threadIdx.y);
//	if (thisThread >= QUEENS * QUEENS * QUEENS * QUEENS)
//		return;
if (blockIdx.x >= QUEENS || blockIdx.y >= QUEENS || threadIdx.x >= QUEENS || threadIdx.y >= QUEENS)
return;

int* queenPos = new int[QUEENS];

queenPos[3] = blockIdx.x;
queenPos[4] = blockIdx.y;
queenPos[5] = threadIdx.x;
queenPos[6] = threadIdx.y;

for (int i = 4; i <= 6; i++) {
for (int j = 3; j < i; j++) {
if ((queenPos[i] - i) == (queenPos[j] - j) || (queenPos[i] + i) == (queenPos[j] + j) || queenPos[i] == queenPos[j]) {
return;
}
}
}
int totalFQP = numFQP[0] / 3;

for (int FQP_number = 0; FQP_number < totalFQP; FQP_number++) {
//	printf("1_%d %d %d %d %d %d %d %d\n", thisThread, blockIdx.x, gridDim.x, blockIdx.y, gridDim.y, threadIdx.x, blockDim.x, threadIdx.y);
//	if (thisThread >= QUEENS * QUEENS * QUEENS * QUEENS)
//		return;

for (int i = 0; i < 3; i++)
queenPos[i] = frontQueensPos[(FQP_number * 3) + i];

bool legal = true;

//if (blockIdx.x == 6 && blockIdx.y == 11 && threadIdx.x == 9 && threadIdx.y == 12)
//	printf("1_%d %d %d %d %d %d %d_%d\n", queenPos[0], queenPos[1], queenPos[2], queenPos[3], queenPos[4], queenPos[5], queenPos[6], totalFQP);

for (int i = 3; i <= 6; i++) {
for (int j = 0; j < 3; j++) {
if ((queenPos[i] - i) == (queenPos[j] - j) || (queenPos[i] + i) == (queenPos[j] + j) || queenPos[i] == queenPos[j]) {
legal = false;
break;
}
}
if (!legal)
break;
}
if (!legal)
continue;

//if (blockIdx.x == 6 && blockIdx.y == 11 && threadIdx.x == 9 && threadIdx.y == 12)
//	printf("1_%d %d %d %d %d %d %d_%d\n", queenPos[0], queenPos[1], queenPos[2], queenPos[3], queenPos[4], queenPos[5], queenPos[6], localResult);

//printf("1_%d %d %d %d %d %d %d\n", thisThread, queenPos[2], blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, data[thisThread]);
//backtrace
int posNow = 7;
queenPos[posNow] = -1;
while (posNow > 6) {
queenPos[posNow] ++;
while (queenPos[posNow] < QUEENS) {
legal = true;
for (int j = posNow - 1; j >= 0; j--) {
if ((queenPos[posNow] - posNow) == (queenPos[j] - j) || (queenPos[posNow] + posNow) == (queenPos[j] + j) || queenPos[posNow] == queenPos[j]) {
legal = false;
break;
}
}
if (!legal)
queenPos[posNow] ++;
else
break;
}
if (queenPos[posNow] < QUEENS) {
if (posNow == (QUEENS - 1)) {
localResult++;
//if (blockIdx.x == 6 && blockIdx.y == 11 && threadIdx.x == 9 && threadIdx.y == 12)
//	printf("2_%d %d %d %d %d %d %d_%d\n", queenPos[7], queenPos[8], queenPos[9], queenPos[10], queenPos[11], queenPos[12], queenPos[13], localResult);
posNow--;
}
else {
posNow++;
queenPos[posNow] = -1;
}
}
else
posNow--;
}
}
//if (blockIdx.x == 6 && blockIdx.y == 11 && threadIdx.x == 9 && threadIdx.y == 12)
//	printf("2.5_%d\n", localResult);
data[thisThread] = localResult;
//if (blockIdx.x == 6 && blockIdx.y == 11 && threadIdx.x == 9 && threadIdx.y == 12)
//	printf("3_%d %d %d %d %d %d\n", thisThread, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, data[thisThread]);
}