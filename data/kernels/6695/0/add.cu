#include "hip/hip_runtime.h"
#include "includes.h"
#define N 10

//Sum Arrays
__global__ void add(int *x, int *y, int *z){
int tID = blockIdx.x;
if (tID < N){
z[tID] = x[tID] + y[tID];
}
}