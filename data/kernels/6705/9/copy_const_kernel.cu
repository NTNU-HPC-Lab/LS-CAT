#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_const_kernel ( float *iptr, const float *cptr ) {
// map from threadIdx/blockIdx to pixel position
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

if(cptr[offset] != 0) iptr[offset] = cptr[offset];
}