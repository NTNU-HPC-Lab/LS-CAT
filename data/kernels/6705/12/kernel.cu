#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(unsigned char *ptr, int ticks){
// map from threadIdx/BlockIdx to pixel positions
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

// now calculate the value at that position
float fx = x - DIM/2;
float fy = y - DIM/2;
float d = sqrtf( fx * fx + fy * fy );

unsigned char grey = (unsigned char) (128.0f + 127.0f * cos(d/10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

ptr[offset*4 + 0] = grey;
ptr[offset*4 + 1] = grey;
ptr[offset*4 + 2] = grey;
ptr[offset*4 + 3] = 255;
}