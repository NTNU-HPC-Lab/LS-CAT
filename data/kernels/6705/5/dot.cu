#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float* a, float* b, float* c) {
__shared__ float cache[threadsPerBlock];
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int cacheIndex = threadIdx.x;

float temp = 0;
while (tid < N){
temp += a[tid] * b[tid];
tid += blockDim.x * gridDim.x;
}

// set the cache values
cache[cacheIndex] = temp;

// synchronize threads in this block
__syncthreads();

// for reductions, threadsPerBlock must be a power of 2
// because of the following code
int i = blockDim.x/2;
while (i != 0){
if (cacheIndex < i)
cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
i /= 2;
}

if (cacheIndex == 0)
c[blockIdx.x] = cache[0];
}