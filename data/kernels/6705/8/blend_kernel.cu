#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blend_kernel( float *outSrc, const float *inSrc ) {
// map from threadIdx/blockIdx to pixel position
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

int left = offset - 1;
int right = offset + 1;
if(x == 0) left++;
if(x == DIM-1) right--;

int top = offset - DIM;
int bottom = offset + DIM;
if(y == 0) top += DIM;
if(y == DIM-1) bottom -= DIM;

outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}