#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertRGBtoGrayScale(uint8_t* src, uint8_t* dst,int width,int height, int channels)
{
int x = threadIdx.x+ blockIdx.x* blockDim.x;
int y = threadIdx.y+ blockIdx.y* blockDim.y;
if(x < width && y < height) {
int grayOffset= y*width + x;// one can think of the RGB image having
int rgbOffset= grayOffset*channels;// CHANNEL times columns than the gray scale
unsigned char r =  src[rgbOffset]; // red value for pixel
unsigned char g = src[rgbOffset+ 2]; // green value for pixel
unsigned char b = src[rgbOffset+ 3]; // blue value for pixel// perform the rescaling and store it// We multiply by floating point constants
dst[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
}
}