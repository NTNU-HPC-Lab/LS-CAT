#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mykernel(void){
//printf("Inside function\n");
printf("Block id: %d\n",blockIdx.x);
printf("Thread id: %d\n",threadIdx.x);
printf("Global id: %d\n",(threadIdx.x + blockIdx.x*blockDim.x));	//blockDim.x is number of threads in a block
}