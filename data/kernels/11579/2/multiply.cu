#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply(int *result, int *A, int *B)
{
/* OLD logic
We have a 3 by 3 grid and each block has 3 threads.
So rows = block x id, cols = block y id
So Indices will be C[block X id][block Y id] = A[block X id][threads 0, 1, 2] * B[threads 0, 1, 2][block y id]
*/
//__shared__ int result[_size*_size] ;
/*result[blockIdx.x*blockDim.x +blockIdx.y] += A[blockIdx.x*blockDim.x + threadIdx.x]*B[blockDim.x*threadIdx.x+blockIdx.y];
printf("C[%d] = A[%d]*B[%d] = %d*%d\n",blockIdx.x*blockDim.x +blockIdx.y, blockIdx.x*blockDim.x + threadIdx.x, blockDim.x*threadIdx.x+blockIdx.y,
A[blockIdx.x*blockDim.x + threadIdx.x],B[blockDim.x*threadIdx.x+blockIdx.y]);
Res[blockIdx.x*blockDim.x +blockIdx.y]= result[blockIdx.x*blockDim.x +blockIdx.y];*/

/* NEW logic
I have 3 blocks and 3 threads. Each thread calculates entry for each position compared to the old one having each thread multiplying one value.
So indices will be result[block x id][thread id] = A[block x id][i]* B[i][thread x id]
*/

for(int i=0; i<_size;i++)
{
result[blockIdx.x*blockDim.x +threadIdx.x] += A[blockIdx.x*blockDim.x+i]*B[blockDim.x*i+threadIdx.x];
}
}