#include "hip/hip_runtime.h"
#include "includes.h"
#define _size 512

__global__ void mul(int *a, int *b, int *c)
{
c[threadIdx.x + blockIdx.x*blockDim.x] = a[threadIdx.x + blockIdx.x*blockDim.x]*b[threadIdx.x + blockIdx.x*blockDim.x];
}