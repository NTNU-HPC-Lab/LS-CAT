#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "remap_reduction.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
uint32_t *d_reduction = NULL;
hipMalloc(&d_reduction, XSIZE*YSIZE);
uint32_t *d_mapping = NULL;
hipMalloc(&d_mapping, XSIZE*YSIZE);
uint32_t *old_d_ij_buf = NULL;
hipMalloc(&old_d_ij_buf, XSIZE*YSIZE);
uint32_t sum_prev_size = XSIZE*YSIZE;
uint32_t prev_size = XSIZE*YSIZE;
uint32_t *new_d_ij_buf = NULL;
hipMalloc(&new_d_ij_buf, XSIZE*YSIZE);
uint32_t new_size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
remap_reduction<<<gridBlock,threadBlock>>>(d_reduction,d_mapping,old_d_ij_buf,sum_prev_size,prev_size,new_d_ij_buf,new_size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
remap_reduction<<<gridBlock,threadBlock>>>(d_reduction,d_mapping,old_d_ij_buf,sum_prev_size,prev_size,new_d_ij_buf,new_size);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
remap_reduction<<<gridBlock,threadBlock>>>(d_reduction,d_mapping,old_d_ij_buf,sum_prev_size,prev_size,new_d_ij_buf,new_size);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}