#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright Douglas Goddard 2016
// Licensed under the MIT license


// shout out to salix alba, you're a wizard mate
// http://stackoverflow.com/a/39862297/1176872
__global__ void map_index_to_prefix( uint8_t *hashes, uint32_t *sort_indices, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
if(t_index < size) {
uint32_t index = sort_indices[t_index];
unsigned char* hash = hashes+index*30*sizeof(unsigned char)+r*3;
uint32_t key = hash[0] << 16 | hash[1] << 8 | hash[2];

uint32_t count = comb_count[key];
uint32_t sum = comb_sum[key];
for(int i=(sum-count); i<sum; i++) {
comb_prefix[i] = key;
}
}
}