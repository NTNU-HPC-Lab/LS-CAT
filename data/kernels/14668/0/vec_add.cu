#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_add(int* A, int* B, int* C, int size) {

int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < size) {
C[index] = A[index] + B[index];
}
}