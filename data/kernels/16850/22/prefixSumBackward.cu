#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSumBackward(float* arr,int step){

int bx = blockIdx.x;
int tx = threadIdx.x;

int BX = blockDim.x;

int i = bx*BX+tx;

int ii = i+1;

if(i >= n || ii > n/float(step)) return;

int temp = arr[ii*step-1];
arr[ii*step-1]	 += arr[ii*step-step/2-1];
arr[ii*step-step/2-1] = temp;

}