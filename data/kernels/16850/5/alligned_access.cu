#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void alligned_access(float* a,int max){
int idx = blockIdx.x*blockDim.x + threadIdx.x;
if (idx >= max) return;
a[idx] = a[idx] + 1;
}