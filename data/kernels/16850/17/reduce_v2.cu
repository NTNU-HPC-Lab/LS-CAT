#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v2(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int BX = blockDim.x; //same as THEAD_MAX
int i  = bx*BX+tx;

__shared__ float S[THEAD_MAX];

S[tx] = i < n ?  in[i] : 0;
__syncthreads();
for(int s=BX/2; s>0 ;s>>=1){
if(tx < s)
S[tx] += S[tx+s];
__syncthreads();
}
if(tx==0)
out[bx] = S[0];
}