#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blur(int* B,int* G,int* R, int* RB,int* RG,int* RR, int* K, int rows, int cols, int krows, int kcols) {

int index = blockIdx.x * 1024 + threadIdx.x;

if (index > rows*cols)
return;

int pixel_row = index/cols ;
int pixel_col = index - pixel_row*cols;

int pr,pc,idx;

int k_sum = 0;
int kr,kc;

int k_center_row = (krows-1)/2;
int k_center_col = (kcols-1)/2;

for(int i=0;i<krows;i++) {
for(int j=0;j<kcols;j++) {

kr = (i - k_center_row);
kc = (j - k_center_col);

pr = pixel_row + kr ;
pc = pixel_col + kc ;

idx = pr*cols + pc;

if (pr >=0 && pr < rows && pc>=0 && pc < cols) {
k_sum += K[kr*kcols + kc];

RB[index] += B[idx]*K[kr*kcols + kc];
RG[index] += G[idx]*K[kr*kcols + kc];
RR[index] += R[idx]*K[kr*kcols + kc];

}
}
}

RB[index] /= k_sum;
RG[index] /= k_sum;
RR[index] /= k_sum;
}