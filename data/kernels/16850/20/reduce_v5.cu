#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void warp_reduce(float* S,int tx){
S[tx] += S[tx + 32]; __syncthreads();
S[tx] += S[tx + 16]; __syncthreads();
S[tx] += S[tx + 8];  __syncthreads();
S[tx] += S[tx + 4];  __syncthreads();
S[tx] += S[tx + 2];  __syncthreads();
S[tx] += S[tx + 1];  __syncthreads();
}
__global__ void reduce_v5(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int i  = bx*(BX*2)+tx;

__shared__ float S[BX];	//Want to have only BX amount of shared mem which is THREAD_MAX in previous

S[tx] = in[i] + in[i+BX]; //Increased part thread activity at start and start only half the threads
__syncthreads();

if(BX >= 1024){                 // Max threads for block in my gpu is 1024
if(tx < 512)
S[tx] += S[tx+512];
__syncthreads();
}

if(BX >= 512){
if(tx < 256)
S[tx] += S[tx+256];
__syncthreads();
}

if(BX >= 256){
if(tx < 128)
S[tx] += S[tx+128];
__syncthreads();
}

if(BX >= 128){
if(tx < 64)
S[tx] += S[tx+64];
__syncthreads();
}

if(tx < WARP_SIZE) {				//WARP_SIZE is 32
warp_reduce(S,tx);				//Unroaling the last warp
}

if(tx==0)
out[bx] = S[0];
}