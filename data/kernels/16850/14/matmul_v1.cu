#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_v1(float* a,float* b,float* c, int n){
// C(nxn) = A(nxn) * B(nxn);

__shared__ float A[TILE_SIZE][TILE_SIZE+1];
__shared__ float B[TILE_SIZE][TILE_SIZE+1];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int i = bx*TILE_SIZE+tx;
int j = by*TILE_SIZE+ty;

A[ty][tx] = A[ty][tx] = 0;
if(i >= n || j >= n) return;

float c_ij = 0;
for(int m=0;m<float(n)/TILE_SIZE;m++){
A[ty][tx] = a[j*n+ m*TILE_SIZE + tx];
B[ty][tx] = b[(m*TILE_SIZE+ty)*n+i];

//		printf("%d %d : %f - %f\n",tx,ty,A[ty][tx],B[ty][tx]);

__syncthreads();

for(int k=0;k<TILE_SIZE;k++)
c_ij += A[ty][k]*B[k][tx];
__syncthreads();
}
c[n*j+i] = c_ij;

}