#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSumForward(float* arr,int step){

int bx = blockIdx.x;
int tx = threadIdx.x;

int BX = blockDim.x;

int i = bx*BX+tx;

int ii = i+1;

if( ii <= n &&  ii > n/float(step)) return;

arr[ii*step-1] += arr[ii*step-step/2-1];

if(step==n && n-1 == ii*step-1) {
arr[ii*step]  = arr[ii*step-1];
arr[ii*step-1]= 0;
}
}