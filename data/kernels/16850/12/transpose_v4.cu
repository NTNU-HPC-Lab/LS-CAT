#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v4(float* a,float* b, int n){

int blockIdx_x = blockIdx.y;
int blockIdx_y = (blockIdx.x+blockIdx.y)%gridDim.x;

int tx = threadIdx.x;
int ty = threadIdx.y;

int bx = blockIdx_x;
int by = blockIdx_y;

int i = bx*BX + tx;
int j = by*BY + ty;

__shared__ float tile[BY][BX+1]; //Very slight modification to avoid bank conflict in shared mem

if(i >= n || j >= n) return;

tile[ty][tx] = a[j*n+i];

__syncthreads();

i = by*BY + tx;
j = bx*BX + ty;

b[j*n+i] = tile[tx][ty];

}