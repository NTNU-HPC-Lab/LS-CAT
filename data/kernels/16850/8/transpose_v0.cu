#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v0(float* a,float* b, int n){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if(i >= n || j >= n) return;

b[n*i+j] = a[n*j+i];

}