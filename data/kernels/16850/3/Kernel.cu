#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel(int* a,int* b,int *c,int n){

int i = blockIdx.x*blockDim.x + threadIdx.x;

__shared__ extern int shared_mem[];
int reg;

if(i>= n) return;

reg = a[i] + b[i];
shared_mem[i] = reg;
c[i] = shared_mem[i];

}