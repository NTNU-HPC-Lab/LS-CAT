#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_v0(float* a,float* b,float* c, int n){
// C(nxn) = A(nxn) * B(nxn);
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if(i >= n || j >= n) return;

float c_ij = 0;
for(int k=0;k<n;k++){
c_ij += a[n*j+k]*b[n*k+i];

//		printf("%d %d %d : %f %f\n",i,j,k,a[n*j+k],b[n*k+i]);

}
c[n*j+i] = c_ij;

}