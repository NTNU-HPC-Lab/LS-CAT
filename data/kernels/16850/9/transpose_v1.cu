#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v1(float* a,float* b, int n){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if(i >= n || j >= n) return;

b[n*j+i] = a[n*i+j];

}