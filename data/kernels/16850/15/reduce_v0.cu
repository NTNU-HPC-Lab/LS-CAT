#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v0(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int BX = blockDim.x; //same as THEAD_MAX
int i  = bx*BX+tx;

__shared__ float S[THEAD_MAX];

S[tx] = i < n ?  in[i] : 0;
__syncthreads();
for(int s=1; s<BX ;s*=2){
if(tx%(2*s)==0)
S[tx] += S[tx+s];
__syncthreads();
}
if(tx==0)
out[bx] = S[0];
}