#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v1(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int BX = blockDim.x; //same as THEAD_MAX
int i  = bx*BX+tx;

__shared__ float S[THEAD_MAX];

S[tx] = i < n ?  in[i] : 0;
__syncthreads();
for(int s=1; s<BX ;s*=2){
int index = 2*s*tx;
if(index < BX)
S[index] += S[index+s];
__syncthreads();
}
if(tx==0)
out[bx] = S[0];
}