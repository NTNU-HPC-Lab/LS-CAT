#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSum(float* arr,int step){

int bx = blockIdx.x;
int tx = threadIdx.x;

int BX = blockDim.x;

int i = bx*BX+tx;

if(i < step) return;

int temp = arr[i-step];
__syncthreads();
arr[i] += temp;
}