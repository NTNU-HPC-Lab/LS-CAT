#include "hip/hip_runtime.h"
#include "includes.h"



// Lenght of each data
__constant__ int gcT_size;
__constant__ int gcP_size;

// Threshold of the SW algorithm
__constant__ int gcThre;

// Data of the query
__constant__ char gcP_seq[1024];

// Cost and Gain
__constant__ int gcMatch;
__constant__ int gcMiss;
__constant__ int gcExtend;
__constant__ int gcBegin;

enum{
Zero,
Diagonal,
Vertical,
Horizon,
};

using namespace std;

__global__ void DP(char* dT_seq, char* dTrace, int* dScore){
// ThreadId = ptn point
int id = threadIdx.x;
// The acid in this thread
char p = gcP_seq[id];
// p-1 row line's value
__shared__ int Hp_1[1024];
__shared__ int Ep_1[1024];
// Temporary
int Hp_1_buf = 0;
int Ep_1_buf = 0;
// t-1 element value
int Ht_1 = 0;
int Ft_1 = 0;
// p-1 t-1 element value
int Ht_1p_1 = 0;
// Initialize
Hp_1[id] = 0;
Ep_1[id] = 0;
// Similar score
int sim = 0;
int point = id * gcT_size - id;
// Culcurate elements
for(int t = -id; t < gcT_size; ++t){
// Control culcurate order
if(t<0){}
// Get similar score
else{
// Compare acids
if(dT_seq[t] == p){sim = gcMatch;}
else{sim = gcMiss;}
}
// SW algorithm
// Culcurate each elements
Ht_1p_1 += sim;	// Diagonal
Ht_1 += gcBegin;	// Horizon (Start)
Ft_1 += gcExtend;	// Horizon (Extend)
Hp_1_buf = Hp_1[id] + gcBegin;	// Vertical (Start)
Ep_1_buf = Ep_1[id] + gcExtend;	// Vertical (Extend)
// Choose the gap score
if(Ht_1 > Ft_1){Ft_1 = Ht_1;}	// Horizon
if(Hp_1_buf > Ft_1){Ep_1_buf = Hp_1_buf;}	// Vertical
// Choose the max score
// Ht_1 is stored the max score
if(Ht_1p_1 > Ep_1_buf){
// Diagonal
if(Ht_1p_1 > Ft_1){
Ht_1 = Ht_1p_1;
dTrace[point] = Diagonal;
}
// Horizon
else{
Ht_1 = Ft_1;
dTrace[point] = Horizon;
}
}
else {
// Vertical
if(Ep_1_buf > Ft_1){
Ht_1 = Ep_1_buf;
dTrace[point] = Vertical;
}
// Horizon
else{
Ht_1 = Ft_1;
dTrace[point] = Horizon;
}
}
// The case 0 is max
if(Ht_1 <= 0){
Ht_1 = 0;
// Set 0 other value
Ft_1 = 0;
Ep_1_buf = 0;
dTrace[point] = Zero;
}
// Hp-1 is next Ht-1p-1
Ht_1p_1 = Hp_1[id];
__syncthreads();
// Set value need next culcurate
// p+1 row line
if(t >= 0){
Hp_1[id + 1] = Ht_1;
Ep_1[id + 1] = Ep_1_buf;
// DEBUG, score check
// dTrace[point] = (char)(Ht_1);
}
if(Ht_1 >= gcThre){
//		printf("Score = %d:\n", Ht_1);
// traceback(dTrace, dT_seq, point-1, t);
if(Ht_1 >= (dScore[t] & 0x0000ffff)){
// Set score and now ptn point
dScore[t] = Ht_1 + (id << 16);
}
}
++point;
__syncthreads();
// for end
}
}