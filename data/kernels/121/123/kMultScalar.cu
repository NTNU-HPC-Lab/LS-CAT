#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len, float scale_targets) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
if (scale_targets == 0) {
for (unsigned int i = idx; i < len; i += numThreads) {
dest[i] = alpha * mat[i];
}
} else {
for (unsigned int i = idx; i < len; i += numThreads) {
dest[i] = scale_targets * dest[i] + alpha * mat[i];
}
}
}