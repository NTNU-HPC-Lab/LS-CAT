#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kShuffleColumns(float* source, float* target, float* indices, int width, int height){
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
float temp1, temp2;
unsigned int column, row, pos1, pos2;

for (unsigned int i = idx; i < height * ((width+1) / 2); i += numThreads) {
column = 2 * (i / height);
row = i % height;
if (column + 1 >= width) {
pos1 = height * (int)indices[column] + row;
target[pos1] = source[pos1];
} else {
pos1 = height * (int)indices[column] + row;
pos2 = height * (int)indices[column + 1] + row;
temp1 = source[pos1];
temp2 = source[pos2];
target[pos2] = temp1;
target[pos1] = temp2;
}
}
}