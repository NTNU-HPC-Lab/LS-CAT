#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] * ((mat[i] > 0) - (mat[i] < 0));
}