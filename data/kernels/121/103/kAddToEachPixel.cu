#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddToEachPixel(float* mat1, float* mat2, float* tgtMat, float mult, unsigned int width, unsigned int height, unsigned int num_pix) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = mat1[i] + mult * mat2[i % height + height * (i / (height * num_pix))];
}
}