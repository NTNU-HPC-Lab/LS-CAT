#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSquashRelu(float* mat, float* target, unsigned int len, float lambda) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = 2 / (1 + __expf(-lambda * mat[i])) - 1;
}