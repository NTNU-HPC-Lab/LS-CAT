#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads)
target[i] = mat[i] * (p[i] >= cutoff) + (1 - mat[i]) * (p[i] < cutoff);
}