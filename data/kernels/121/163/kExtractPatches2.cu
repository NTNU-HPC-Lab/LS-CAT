#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches2(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
int image_id = blockIdx.z % num_images;
int color = blockIdx.z / num_images;
int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
int dest_row = blockIdx.y * blockDim.y + threadIdx.y;

if (dest_col < patch_width && dest_row < patch_height) {
int source_row = int(height_offset[image_id]) + dest_row;
int source_col = int(width_offset[image_id]) + dest_col;
source_col = (flip[image_id] > 0.5) ? (img_width - source_col - 1) : source_col;
unsigned long dest_index = image_id + num_images * (dest_col  + patch_width * (dest_row + patch_height * color));
unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));
patches[dest_index] = images[source_index];
}
}