#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float sigmoid(float x) {
return 1.0f / (1.0f + __expf(-x));
}
__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = sigmoid(mat[i]);
}