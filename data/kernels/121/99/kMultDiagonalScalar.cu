#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width; i += numThreads) {
tgtMat[width*i + i] = mat[width*i + i] * val;
}
}