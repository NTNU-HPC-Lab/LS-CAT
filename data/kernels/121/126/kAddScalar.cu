#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = a[i] + alpha;
}
}