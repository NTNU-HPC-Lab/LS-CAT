#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads)
target[i] = -mat[i] * __logf(p[i] + tiny) - (1 - mat[i]) * __logf(1 - p[i] + tiny);
}