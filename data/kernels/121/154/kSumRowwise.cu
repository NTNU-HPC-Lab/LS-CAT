#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
extern __shared__ float sum_vals[];
const int row = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
if (row < height) {
float sum = 0;
float *data = mat + row;
for (unsigned int i = 0; i < width; i++) sum += data[i*height];
__syncthreads();
target[row] = p * target[row] + mult * sum;
}
}