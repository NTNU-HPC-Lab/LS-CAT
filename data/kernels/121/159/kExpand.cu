#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExpand(float* source, float* indices, float* target, int height, int width, int target_width){
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < target_width*height; i += numThreads) {
const int pos = height * (int)indices[i / height] + i % height;
target[i] = (pos < height * width)? source[pos] : 1.0/0.0 - 1.0/0.0;
}
}