#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBesselRatioActivation(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) {
float r = mat[i];
target[i] = cyl_bessel_i1f(r) / cyl_bessel_i0f(r);
}
}