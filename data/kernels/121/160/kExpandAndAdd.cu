#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2){
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width*height; i += numThreads) {
const int pos = height * (int)indices[i / height] + i % height;
target[i] = (pos < height * width2)? source[i] + mult * mat[pos] : 1.0/0.0 - 1.0/0.0;
}
}