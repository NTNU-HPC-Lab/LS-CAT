#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSwapColumns(float* source, float* target, float* indices1, float* indices2, int cols, int width, int height){
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
float temp;
unsigned int column, row, source_pos, target_pos;
for (unsigned int i = idx; i < height * cols; i += numThreads) {
column = i / height;
row = i % height;
source_pos = height * (int)indices1[column] + row;
target_pos = height * (int)indices2[column] + row;
temp = source[source_pos];
source[source_pos] = target[target_pos];
target[target_pos] = temp;
}
}