#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = mat[i] + mult * vec[i % height];
}
}