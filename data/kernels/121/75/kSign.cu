#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSign(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] ? copysignf(1., mat[i]) : 0;
}