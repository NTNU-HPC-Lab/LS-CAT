#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width * height; i += numThreads) {
target[i] = mat[i] - ((int)labels[i / height] == i % height ? 1 : 0);
}
}