#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches3(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {

int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
int dest_row = blockIdx.y * blockDim.y + threadIdx.y;
if (dest_col < patch_width && dest_row < patch_height) {
for (unsigned int b = blockIdx.z; b < num_colors * num_images; b += gridDim.z) {
int color    = b % num_colors;
int image_id = b / num_colors;
int source_row = int(height_offset[image_id]) + dest_row;
int source_col = int(width_offset[image_id]) + dest_col;
source_col = (flip[image_id] > 0.5) ? (img_width - source_col - 1) : source_col;
unsigned long source_index = source_col + img_width   * (source_row + img_height   * (color + num_colors * image_id));
unsigned long dest_index   = dest_col   + patch_width * (dest_row   + patch_height * (color + num_colors * image_id));
__syncthreads();
patches[dest_index] = images[source_index];
}
}
}