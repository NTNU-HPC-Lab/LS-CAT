#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif






}
__global__ void tensor_5d_equals (const int n, const int c, const int d, const int h, const int w, const DTYPE* x, const int offset_x, const int n_x, const int c_x, const int d_x, const int h_x, const int w_x, const DTYPE* y, const int offset_y, const int n_y, const int c_y, const int d_y, const int h_y, const int w_y, int* eq_flag) {
const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
const int gid_d = blockIdx.z * blockDim.z + threadIdx.z;
const bool valid = (gid_n < n) && (gid_c < c) && (gid_d < d);
if (valid) {
const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_d * d_x;
const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_d * d_y;
for (int i = 0; i < h; i++) {
for (int j = 0; j < w; j++) {
if (x[ix + i * h_x + j * w_x] != y[iy + i * h_y + j * w_y]){
eq_flag[0]++;
}
}
};
}
}