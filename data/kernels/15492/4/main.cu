#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tensor_5d_equals.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int n = 1;
const int c = 1;
const int d = 1;
const int h = 1;
const int w = 1;
const DTYPE *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
const int offset_x = 1;
const int n_x = 1;
const int c_x = 1;
const int d_x = 1;
const int h_x = 1;
const int w_x = 1;
const DTYPE *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
const int offset_y = 1;
const int n_y = 1;
const int c_y = 1;
const int d_y = 1;
const int h_y = 1;
const int w_y = 1;
int *eq_flag = NULL;
hipMalloc(&eq_flag, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
tensor_5d_equals<<<gridBlock,threadBlock>>>(n,c,d,h,w,x,offset_x,n_x,c_x,d_x,h_x,w_x,y,offset_y,n_y,c_y,d_y,h_y,w_y,eq_flag);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
tensor_5d_equals<<<gridBlock,threadBlock>>>(n,c,d,h,w,x,offset_x,n_x,c_x,d_x,h_x,w_x,y,offset_y,n_y,c_y,d_y,h_y,w_y,eq_flag);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
tensor_5d_equals<<<gridBlock,threadBlock>>>(n,c,d,h,w,x,offset_x,n_x,c_x,d_x,h_x,w_x,y,offset_y,n_y,c_y,d_y,h_y,w_y,eq_flag);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}