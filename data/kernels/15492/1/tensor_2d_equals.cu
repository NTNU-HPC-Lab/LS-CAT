#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif






}
__global__ void tensor_2d_equals (const int n, const int c, const DTYPE* x, const int offset_x, const int n_x, const int c_x, const DTYPE* y, const int offset_y, const int n_y, const int c_y, int* eq_flag) {
const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_n < n) && (gid_c < c);
if (valid) {
const int ix = offset_x + gid_n * n_x + gid_c * c_x;
const int iy = offset_y + gid_n * n_y + gid_c * c_y;
if (x[ix] != y[iy]){
eq_flag[0]++;
}
}
}