#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif






}
__global__ void tensor_1d_equals (const int n, const DTYPE* x, const int offset_x, const int stride_x, const DTYPE* y, const int offset_y, const int stride_y, int* eq_flag) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
const int ix = offset_x + gid * stride_x;
const int iy = offset_y + gid * stride_y;
if (x[ix] != y[iy]) {
eq_flag[0]++;
}
}
}