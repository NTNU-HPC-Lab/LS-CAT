#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif






}
__global__ void tensor_4d_equals (const int n, const int c, const int h, const int w, const DTYPE* x, const int offset_x, const int n_x, const int c_x, const int h_x, const int w_x, const DTYPE* y, const int offset_y, const int n_y, const int c_y, const int h_y, const int w_y, int* eq_flag) {
const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
const int gid_h = blockIdx.z * blockDim.z + threadIdx.z;
const bool valid = (gid_n < n) && (gid_c < c) && (gid_h < h);
if (valid) {
const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_h * h_x;
const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_h * h_y;
for (int i = 0; i < w; i++) {
if (x[ix + i * w_x] != y[iy + i * w_y]){
eq_flag[0]++;
}
};
}
}