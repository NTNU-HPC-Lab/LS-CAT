#include "hip/hip_runtime.h"
#include "includes.h"

#define C0  0
#define CZ1 1
#define CX1 2
#define CY1 3
#define CZ2 4
#define CX2 5
#define CY2 6
#define CZ3 7
#define CX3 8
#define CY3 9
#define CZ4 10
#define CX4 11
#define CY4 12

__global__ void inject_Source(int id, int ii, float *p, float *_tableS, float *_sourceV, int *_locsS, int _dir, int _jt, int _ntSrc){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
p[_locsS[ix]]+=(float)_dir/_jt * (
_tableS[ii + 0]*_sourceV[_ntSrc*ix+id]+
_tableS[ii + 1]*_sourceV[_ntSrc*ix+id+1]+
_tableS[ii + 2]*_sourceV[_ntSrc*ix+id+2]+
_tableS[ii + 3]*_sourceV[_ntSrc*ix+id+3]+
_tableS[ii + 4]*_sourceV[_ntSrc*ix+id+4]+
_tableS[ii + 5]*_sourceV[_ntSrc*ix+id+5]+
_tableS[ii + 6]*_sourceV[_ntSrc*ix+id+6]+
_tableS[ii + 7]*_sourceV[_ntSrc*ix+id+7]
);
}