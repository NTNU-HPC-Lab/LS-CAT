#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "inject_Source.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int id = 1;
int ii = 1;
float *p = NULL;
hipMalloc(&p, XSIZE*YSIZE);
float *_tableS = NULL;
hipMalloc(&_tableS, XSIZE*YSIZE);
float *_sourceV = NULL;
hipMalloc(&_sourceV, XSIZE*YSIZE);
int *_locsS = NULL;
hipMalloc(&_locsS, XSIZE*YSIZE);
int _dir = 1;
int _jt = 1;
int _ntSrc = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
inject_Source<<<gridBlock,threadBlock>>>(id,ii,p,_tableS,_sourceV,_locsS,_dir,_jt,_ntSrc);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
inject_Source<<<gridBlock,threadBlock>>>(id,ii,p,_tableS,_sourceV,_locsS,_dir,_jt,_ntSrc);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
inject_Source<<<gridBlock,threadBlock>>>(id,ii,p,_tableS,_sourceV,_locsS,_dir,_jt,_ntSrc);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}