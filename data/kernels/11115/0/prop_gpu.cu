#include "hip/hip_runtime.h"
#include "includes.h"

#define C0  0
#define CZ1 1
#define CX1 2
#define CY1 3
#define CZ2 4
#define CX2 5
#define CY2 6
#define CZ3 7
#define CX3 8
#define CY3 9
#define CZ4 10
#define CX4 11
#define CY4 12

__global__ void prop_gpu(float *p0, float *p1, float *vel, float *coeffs, int _nx, int _ny, int _nz, int _n12){

printf("At the gpu kernel\n");
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int ii  = row * _nx + col;

if (col >= 4 && col < _nz - 4 && row >= 4 && row < _ny - 4){
for(int z = 4; z < _nz-4; z++){
p0[ii]=vel[ii]*
(
coeffs[0]*p1[ii]
+coeffs[1]*(p1[ii-1]+p1[ii+1])+
+coeffs[2]*(p1[ii-2]+p1[ii+2])+
+coeffs[3]*(p1[ii-3]+p1[ii+3])+
+coeffs[4]*(p1[ii-4]+p1[ii+4])+
+coeffs[5]*(p1[ii-_nx]+p1[ii+_nx])+
+coeffs[6]*(p1[ii-2*_nx]+p1[ii+2*_nx])+
+coeffs[7]*(p1[ii-3*_nx]+p1[ii+3*_nx])+
+coeffs[8]*(p1[ii-4*_nx]+p1[ii+4*_nx])+
+coeffs[9]*(p1[ii-1*_n12]+p1[ii+1*_n12])+
+coeffs[10]*(p1[ii-2*_n12]+p1[ii+2*_n12])+
+coeffs[11]*(p1[ii-3*_n12]+p1[ii+3*_n12])+
+coeffs[12]*(p1[ii-4*_n12]+p1[ii+4*_n12])
)
+p1[ii]+p1[ii]-p0[ii];

ii = ii + _n12;
}

}
}