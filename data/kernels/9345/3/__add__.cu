#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __add__(int *array, int *size) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx > *size) return;

int temp = 0;
int before = (idx + 1) % *size;
int after = idx - 1;
if (after < 0) after = *size - 1;


temp += array[idx];
temp += array[before];
temp += array[after];

__syncthreads(); // Barrera...
array[idx] = temp;
}