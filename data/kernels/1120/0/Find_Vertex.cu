#include "hip/hip_runtime.h"
#include "includes.h"
//  Author: Jose F. Martinez Rivera
//  Course: ICOM4036 - 040
//	Professor: Wilson Rivera Gallego
// 	Assignment 2 - CUDA Implementation



#define V 8
#define E 11
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;
//
//Represents an edge or path between Vertices
typedef struct
{
int u;
int v;

} Edge;

//Represents a Vertex
typedef struct
{
int title;
boolean visited;

} Vertex;


//Finds the weight of the path from vertex u to vertex v
__device__ __host__ int findEdge(Vertex u, Vertex v, Edge *edges, int *weights)
{

int i;
for(i = 0; i < E; i++)
{

if(edges[i].u == u.title && edges[i].v == v.title)
{
return weights[i];
}
}

return MAX_WEIGHT;

}
__global__ void Find_Vertex(Vertex *vertices, Edge *edges, int *weights, int *length, int *updateLength)
{

int u = threadIdx.x;


if(vertices[u].visited == FALSE)
{


vertices[u].visited = TRUE;


int v;

for(v = 0; v < V; v++)
{
//Find the weight of the edge
int weight = findEdge(vertices[u], vertices[v], edges, weights);

//Checks if the weight is a candidate
if(weight < MAX_WEIGHT)
{
//If the weight is shorter than the current weight, replace it
if(updateLength[v] > length[u] + weight)
{
updateLength[v] = length[u] + weight;
}
}
}

}

}