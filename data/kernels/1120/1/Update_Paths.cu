#include "hip/hip_runtime.h"
#include "includes.h"
//  Author: Jose F. Martinez Rivera
//  Course: ICOM4036 - 040
//	Professor: Wilson Rivera Gallego
// 	Assignment 2 - CUDA Implementation



#define V 8
#define E 11
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;
//
//Represents an edge or path between Vertices
typedef struct
{
int u;
int v;

} Edge;

//Represents a Vertex
typedef struct
{
int title;
boolean visited;

} Vertex;


//Finds the weight of the path from vertex u to vertex v
__global__ void Update_Paths(Vertex *vertices, int *length, int *updateLength)
{
int u = threadIdx.x;
if(length[u] > updateLength[u])
{

length[u] = updateLength[u];
vertices[u].visited = FALSE;
}

updateLength[u] = length[u];


}