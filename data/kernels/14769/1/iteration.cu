#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCK_SIZE_X = 26;
const int BLOCK_SIZE_Y = 26;
const float w1 = 4.0/9.0, w2 = 1.0/9.0, w3 = 1.0/36.0;
const float Amp2 = 0.1, Width = 10, omega = 1;





__global__ void iteration(float* f_d, int ArraySizeX, int ArraySizeY)
{
int i;
int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x*(BLOCK_SIZE_X-2);
int by = blockIdx.y*(BLOCK_SIZE_Y-2);
int x = tx + bx;
int y = ty + by;
register float n,ux,uy,uxx,uyy,uxy,usq,Fx,Fy,Fxx,Fyy,Fxy,Fsq;
__shared__ float f_sh[BLOCK_SIZE_X][BLOCK_SIZE_Y][9];
//  __shared__ float f_p[BLOCK_SIZE_X][BLOCK_SIZE_Y][9];

for(i=0;i<9;i++)
f_sh[tx][ty][i]=f_d[x*ArraySizeY*9+y*9+ i];

__syncthreads();

n=f_sh[tx][ty][0]+f_sh[tx][ty][1]+f_sh[tx][ty][2]+f_sh[tx][ty][3]+f_sh[tx][ty][4]+f_sh[tx][ty][5]+f_sh[tx][ty][6]+f_sh[tx][ty][7]+f_sh[tx][ty][8];
ux=f_sh[tx][ty][1]-f_sh[tx][ty][2]+f_sh[tx][ty][5]-f_sh[tx][ty][6]-f_sh[tx][ty][7]+f_sh[tx][ty][8];
uy=f_sh[tx][ty][3]-f_sh[tx][ty][4]+f_sh[tx][ty][5]+f_sh[tx][ty][6]-f_sh[tx][ty][7]-f_sh[tx][ty][8];
ux/=n;
uy/=n;
uxx=ux*ux;
uyy=uy*uy;
uxy=2*ux*uy;
usq=uxx+uyy;
// implement the forcing terms and perform collision step
Fx=0;//Amp*sin(y*2*M_PI/cols);
Fy=0;
Fxx=2*n*Fx*ux;
Fyy=2*n*Fy*uy;
Fxy=2*n*(Fx*uy+Fy*ux);
Fsq=Fxx+Fyy;
Fx*=n;
Fy*=n;

f_sh[tx][ty][0]+=omega*(w1*n*(1-1.5*usq)-f_sh[tx][ty][0])-w1*1.5*Fsq;
f_sh[tx][ty][1]+=omega*(w2*n*(1+3*ux+4.5*uxx -1.5*usq)-f_sh[tx][ty][1])+w2*(3*Fx+4.5*Fxx-1.5*Fsq);
f_sh[tx][ty][2]+=omega*(w2*n*(1-3*ux+4.5*uxx -1.5*usq)-f_sh[tx][ty][2])+w2*(-3*Fx+4.5*Fxx-1.5*Fsq);
f_sh[tx][ty][3]+=omega*(w2*n*(1+3*uy+4.5*uyy -1.5*usq)-f_sh[tx][ty][3])+w2*(3*Fy+4.5*Fyy-1.5*Fsq);
f_sh[tx][ty][4]+=omega*(w2*n*(1-3*uy+4.5*uyy -1.5*usq)-f_sh[tx][ty][4])+w2*(-3*Fy+4.5*Fyy-1.5*Fsq);
f_sh[tx][ty][5]+=omega*(w3*n*(1+3*(ux+uy)+4.5*(uxx+uxy+uyy)-1.5*usq)-f_sh[tx][ty][5])+w3*(3*(Fx+Fy)+4.5*(Fxx+Fxy+Fyy)-1.5*Fsq);
f_sh[tx][ty][6]+=omega*(w3*n*(1+3*(-ux+uy)+4.5*(uxx-uxy+uyy)-1.5*usq)-f_sh[tx][ty][6])+w3*(3*(-Fx+Fy)+4.5*(Fxx-Fxy+Fyy)-1.5*Fsq);
f_sh[tx][ty][7]+=omega*(w3*n*(1+3*(-ux-uy)+4.5*(uxx+uxy+uyy)-1.5*usq)-f_sh[tx][ty][7])+w3*(3*(-Fx-Fy)+4.5*(Fxx+Fxy+Fyy)-1.5*Fsq);
f_sh[tx][ty][8]+=omega*(w3*n*(1+3*(ux-uy)+4.5*(uxx-uxy+uyy)-1.5*usq)-f_sh[tx][ty][8])+w3*(3*(Fx-Fy)+4.5*(Fxx-Fxy+Fyy)-1.5*Fsq);
__syncthreads();

//perfom stream step
if(tx>0 && tx<BLOCK_SIZE_X-1 && ty>0 && ty<BLOCK_SIZE_Y-1) {
f_d[x*ArraySizeY*9+y*9] = f_sh[tx][ty][0];//+omega*(w1*n*(1-1.5*usq)-f_sh[tx][ty][0])-w1*1.5*Fsq;
f_d[x*ArraySizeY*9+y*9+2] = f_sh[tx+1][ty][2];//+omega*(w2*n*(1-3*ux+4.5*uxx -1.5*usq)-f_sh[tx+1][ty][2])+w2*(-3*Fx+4.5*Fxx-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+1] = f_sh[tx-1][ty][1];//+omega*(w2*n*(1+3*ux+4.5*uxx -1.5*usq)-f_sh[tx-1][ty][1])+w2*(3*Fx+4.5*Fxx-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+4] = f_sh[tx][ty+1][4];//+omega*(w2*n*(1-3*uy+4.5*uyy -1.5*usq)-f_sh[tx][ty+1][4])+w2*(-3*Fy+4.5*Fyy-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+3] = f_sh[tx][ty-1][3];//+omega*(w2*n*(1+3*uy+4.5*uyy -1.5*usq)-f_sh[tx][ty-1][3])+w2*(3*Fy+4.5*Fyy-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+7] = f_sh[tx+1][ty+1][7];//+omega*(w3*n*(1+3*(-ux-uy)+4.5*(uxx+uxy+uyy)-1.5*usq)-f_sh[tx+1][ty+1][7])+w3*(3*(-Fx-Fy)+4.5*(Fxx+Fxy+Fyy)-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+5] = f_sh[tx-1][ty-1][5];//+omega*(w3*n*(1+3*(ux+uy)+4.5*(uxx+uxy+uyy)-1.5*usq)-f_sh[tx-1][ty-1][5])+w3*(3*(Fx+Fy)+4.5*(Fxx+Fxy+Fyy)-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+6] = f_sh[tx+1][ty-1][6];//+omega*(w3*n*(1+3*(-ux+uy)+4.5*(uxx-uxy+uyy)-1.5*usq)-f_sh[tx+1][ty-1][6])+w3*(3*(-Fx+Fy)+4.5*(Fxx-Fxy+Fyy)-1.5*Fsq);
f_d[x*ArraySizeY*9+y*9+8] = f_sh[tx-1][ty+1][8];//+omega*(w3*n*(1+3*(ux-uy)+4.5*(uxx-uxy+uyy)-1.5*usq)-f_sh[tx-1][ty+1][8])+w3*(3*(Fx-Fy)+4.5*(Fxx-Fxy+Fyy)-1.5*Fsq);

}

__syncthreads();
// apply periodi boundary conditions;
if(x == 0)
for(i = 0;i<9;i++)
f_d[x*ArraySizeY*9+y*9+i] = f_d[(ArraySizeX-2)*ArraySizeY*9+y*9+i];
if(x==ArraySizeX-1)
for(i=0;i<9;i++)
f_d[x*ArraySizeY*9+y*9+i] = f_d[ArraySizeY*9+y*9+i];
if(y == 0)
for(i = 0;i<9;i++)
f_d[x*ArraySizeY*9+y*9+i] = f_d[x*ArraySizeY*9+(ArraySizeY-2)*9+i];
if(y == ArraySizeY-1)
for(i =0;i<9;i++)
f_d[x*ArraySizeY*9 +y*9 +i] = f_d[x*ArraySizeY*9+9+i];

//   __syncthreads();
}