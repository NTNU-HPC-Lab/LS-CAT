#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCK_SIZE_X = 26;
const int BLOCK_SIZE_Y = 26;
const float w1 = 4.0/9.0, w2 = 1.0/9.0, w3 = 1.0/36.0;
const float Amp2 = 0.1, Width = 10, omega = 1;





__global__ void Denrho(float* u_d, float* f_d, int ArraySizeX, int ArraySizeY)
{
int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x*(BLOCK_SIZE_X-2);
int by = blockIdx.y*(BLOCK_SIZE_Y-2);
int x = tx + bx;
int y = ty + by;
u_d[x*ArraySizeY+y] = 0;
for (int i=0;i<9;i++)
u_d[x*ArraySizeY+y] += (float)f_d[x*ArraySizeY*9+y*9+i];

__syncthreads();
}