#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_Path(int *Md, const int Width, const int k)
{
//2 Thread ID
int ROW = blockIdx.x;
int COL = threadIdx.x;


if (Md[ROW * Width + COL] > Md[ROW * Width + k] + Md[k * Width + COL])
Md[ROW * Width + COL] = Md[ROW * Width + k] + Md[k * Width + COL];
}