#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_calculate_sum(double * dev_array_sums, unsigned int array_size, double * dev_block_sums) {
//
// sum of input array
//
__shared__ double shared_sum[BLOCK_SIZE];

// each thread loads one element from global to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < array_size)
{
shared_sum[tid] = dev_array_sums[i];
}
else
{
shared_sum[tid] = 0;
}
//synchronize the local threads writing to the local memory cache
__syncthreads();
// do reduction in shared memory
for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
if (tid < s) {
shared_sum[tid] += shared_sum[tid + s];
}
__syncthreads();
}
// write result for this block to global mem
if (tid == 0)
{
dev_block_sums[blockIdx.x] = shared_sum[0];
}
}