#include "hip/hip_runtime.h"
#include "includes.h"

#define N 33 * 1024
#define threadsPerBlock 256
#define blocksPerGrid (N + threadsPerBlock - 1) / threadsPerBlock
#define RADIUS 2
// Signal/image element type
typedef int element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     N      - length of the signal



//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     N      - length of the signal
__global__ void _medianfilter(const element* signal, element* result)
{
__shared__ element cache[threadsPerBlock + 2 * RADIUS];
element window[5];
int gindex = threadIdx.x + blockDim.x * blockIdx.x;
int lindex = threadIdx.x + RADIUS;
// Reads input elements into shared memory
cache[lindex] = signal[gindex];
if (threadIdx.x < RADIUS)
{
cache[lindex - RADIUS] = signal[gindex - RADIUS];
cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
}
__syncthreads();
for (int j = 0; j < 2 * RADIUS + 1; ++j)
window[j] = cache[threadIdx.x + j];
// Orders elements (only half of them)
for (int j = 0; j < RADIUS + 1; ++j)
{
// Finds position of minimum element
int min = j;
for (int k = j + 1; k < 2 * RADIUS + 1; ++k)
if (window[k] < window[min])
min = k;
// Puts found minimum element in its place
const element temp = window[j];
window[j] = window[min];
window[min] = temp;
}
// Gets result - the middle element
result[gindex] = window[RADIUS];
}