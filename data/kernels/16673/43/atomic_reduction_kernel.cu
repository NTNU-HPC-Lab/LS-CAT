#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomic_reduction_kernel(float *data_out, float *data_in, int size)
{
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

atomicAdd(&data_out[0], data_in[idx_x]);
}