#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void child_kernel(int *data, int seed)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

atomicAdd(&data[idx], seed);
}