#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
row += 1;

float sum = 0.f;
for (int i = 0; i < K; ++i)
sum += A[row * K + i] * B[i * K + col];

C[row * M + col] = alpha * sum + beta * C[row * M + col];
}