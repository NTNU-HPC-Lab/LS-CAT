#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateBodyForce(float4 *p, float4 *v, float dt, int n) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < n) {
float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

for (int tile = 0; tile < gridDim.x; tile++) {
__shared__ float3 shared_position[BLOCK_SIZE];
float4 temp_position = p[tile * blockDim.x + threadIdx.x];
shared_position[threadIdx.x] = make_float3(temp_position.x, temp_position.y, temp_position.z);
__syncthreads(); //synchronoze to make sure all tile data is available in shared memory

for (int j = 0; j < BLOCK_SIZE; j++) {
float dx = shared_position[j].x - p[i].x;
float dy = shared_position[j].y - p[i].y;
float dz = shared_position[j].z - p[i].z;
float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
float invDist = rsqrtf(distSqr);
float invDist3 = invDist * invDist * invDist;

Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
}
__syncthreads(); // synchrnize before looping to other time
} //tile loop ends here

v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
} //if ends here
}