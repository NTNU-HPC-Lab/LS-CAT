#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel_1(float* g_out, float* g_in, unsigned int size)
{
unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

extern __shared__ float s_data[];

s_data[threadIdx.x] = (idx_x < size) ? g_in[idx_x] : 0.f;

__syncthreads();

// do reduction
// interleaved addressing
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
{
int index = 2 * stride * threadIdx.x;

if (index < blockDim.x)
s_data[index] += s_data[index + stride];

__syncthreads();
}

if (threadIdx.x == 0)
g_out[blockIdx.x] = s_data[0];
}