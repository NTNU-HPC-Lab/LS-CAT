#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduction_kernel(float *data_out, float *data_in, int stride, int size)
{
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

if (idx_x + stride < size) {
data_out[idx_x] += data_in[idx_x + stride];
}
}