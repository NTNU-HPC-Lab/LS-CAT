#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void idx_print()
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int warp_idx = threadIdx.x / warpSize;
int lane_idx = threadIdx.x & (warpSize - 1);

if ((lane_idx & (warpSize/2 - 1)) == 0)
//  thread, block, warp, lane"
printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx);
}