#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pack_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;

if (idx >= length) return;

if (d_predicates[idx] != 0.f)
{
// address
int address = d_scanned[idx] - 1;

// gather
d_output[address] = d_input[idx];
}
}