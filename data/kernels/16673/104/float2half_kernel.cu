#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float2half_kernel(half *out, float *in)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

out[idx] = __float2half(in[idx]);
}