#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

float element_c = 0.f;
for (int e = 0; e < K; e++)
element_c += A[row * K + e] * B[e * K + col];

C[row * N + col] = alpha * element_c + beta * C[row * N + col];
}