#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void split_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;

if (idx >= length) return;

if (d_predicates[idx] != 0.f)
{
// address
int address = d_scanned[idx] - 1;

// split
d_output[idx] = d_input[address];
}
}