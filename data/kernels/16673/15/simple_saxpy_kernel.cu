#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_saxpy_kernel(float *y, const float* x, const float alpha, const float beta)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

y[idx] = alpha * x[idx] + beta;
}