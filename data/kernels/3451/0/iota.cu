#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iota(int const size, int *data, int const value)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < size)
data[idx] = idx + value;
}