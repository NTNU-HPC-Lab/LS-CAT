#include "hip/hip_runtime.h"
#include "includes.h"


// Bodies_input array contains position [0,1], velocity [2,3], mass [4]
// Bodies_output array contains position [0,1], velocity [2,3], mass [4]; mass is not used here

__global__ void forces_and_step(double *bodies_input, double *bodies_output, unsigned int count, double dt, uint16_t bods_per_thread)
{
unsigned int index = ((blockIdx.x * blockDim.x) + threadIdx.x) * 5 * bods_per_thread;
//unsigned int index = ((blockIdx.x * blockDim.x) + threadIdx.x) * 5;

for (int b = 0; b < bods_per_thread; b++)
{
// If there are more threads than data discard the extra computations to stay in defined memory
if (index < count * 5)
{
// Calculate force for this particle
double fx = 0.0;
double fy = 0.0;
for (uint32_t i = 0; i < count * 5; i += 5)
{
double dir_x = bodies_input[i] - bodies_input[index];
double dir_y = bodies_input[i + 1] - bodies_input[index + 1];
// Make sure there is no division by zero
if (dir_x == 0.0 && dir_y == 0.0)
continue;
fx += G_CONSTANT * bodies_input[i + 4] * bodies_input[index + 4] * dir_x
/ pow(sqrt(dir_x * dir_x + dir_y * dir_y), 3.0);
fy += G_CONSTANT * bodies_input[i + 4] * bodies_input[index + 4] * dir_y
/ pow(sqrt(dir_x * dir_x + dir_y * dir_y), 3.0);
}

// Integration
bodies_output[index + 2] = bodies_input[index + 2] + (fx / bodies_input[index + 4]) * dt;
bodies_output[index + 3] = bodies_input[index + 3] + (fy / bodies_input[index + 4]) * dt;
bodies_output[index] = bodies_input[index] + bodies_output[index + 2] * dt;
bodies_output[index + 1] = bodies_input[index + 1] + bodies_output[index + 3] * dt;
bodies_output[index + 4] = bodies_input[index + 4];
}
index += 5;
}
}