#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gHighwayBackward(float* out1, float* out2, float* outt, const float* in1, const float* in2, const float* t, const float* adj, size_t length) {
for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
if(index < length) {
float sigma = stableSigmoid(t[index]);
out1[index] = sigma * adj[index];
out2[index] = (1.f - sigma) * adj[index];
outt[index]
= sigma * (1.f - sigma) * (in1[index] - in2[index]) * adj[index];
}
}
}