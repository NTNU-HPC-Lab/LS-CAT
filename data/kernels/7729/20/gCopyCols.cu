#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gCopyCols(float* out, const float* in, size_t rows, size_t colsIn, const size_t* sourceColIdx, size_t colsOut) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
const float* rowIn = in + j * colsIn;
float* rowOut = out + j * colsOut;

for(int tid = 0; tid < colsOut; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < colsOut)
rowOut[i] = rowIn[sourceColIdx[i]];
}
}
}
}