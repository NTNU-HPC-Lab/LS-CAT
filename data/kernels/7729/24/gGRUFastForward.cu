#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gGRUFastForward(float* out, const float* state, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols, bool final) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float m = !mask || mask[j];
float* rowOut = out + j * cols;
const float* rowState = state + j * cols;

const float* xWrow = xW + j * cols * 3;
const float* sUrow = sU + j * cols * 3;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
float r = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);

int k = i + cols;

float z = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

int l = i + 2 * cols;
float h;
if(final)
h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
else
h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

float out = (1.0f - z) * h + z * rowState[i];
rowOut[i] = m * out + (1 - m) * rowState[i];
}
}
}
}
}