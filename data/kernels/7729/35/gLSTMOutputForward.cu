#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gLSTMOutputForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, size_t rows, size_t cols) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float* rowOut = out + j * cols;
const float* rowCell = cell + j * cols;

const float* xWrow = xW + j * cols * 4;
const float* sUrow = sU + j * cols * 4;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
int k = i + 3 * cols;
float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

rowOut[i] = go * tanhf(rowCell[i]);
}
}
}
}
}