#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gJoin2(float* out, size_t rowBatch, size_t cols, const float* in1, size_t inStride1, const float* in2, size_t inStride2) {
int outStride = inStride1 + inStride2;
int rows = rowBatch * outStride;

for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float* rowOut = out + j * cols;

int curBatch = j / outStride;
int curPos = j % outStride;

int jIn1 = (curBatch * inStride1) + curPos;
int jIn2 = (curBatch * inStride2) + curPos - inStride1;

const float* rowIn1 = in1 + jIn1 * cols;
const float* rowIn2 = in2 + jIn2 * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
if(curPos < inStride1)
rowOut[i] = rowIn1[i];
else
rowOut[i] = rowIn2[i];
}
}
}
}
}