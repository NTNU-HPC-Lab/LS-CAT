#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gPasteRows(float* out, const float* in, size_t cols, const size_t* targetRowIdx, size_t rows) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
size_t dstId = targetRowIdx[j];
size_t srcId = j;

float* rowOut = out + dstId * cols;
const float* rowIn = in + srcId * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols)
atomicAdd(rowOut + i, rowIn[i]);
}
}
}
}