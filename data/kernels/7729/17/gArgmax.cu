#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gArgmax(float* out, const float* data, size_t rows, size_t cols) {
size_t row = blockIdx.x;
size_t startInd = row * cols;
float maxScore = -99999;
size_t maxInd;
for(size_t col = 0; col < cols; ++col) {
size_t ind = startInd + col;
float score = data[ind];
if(score > maxScore) {
maxScore = score;
maxInd = col;
}
}
out[row] = maxInd;
}