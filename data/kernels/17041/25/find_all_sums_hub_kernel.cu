#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_all_sums_hub_kernel(int* hub, int nhub, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result){
int x = blockIdx.x * blockDim.x + threadIdx.x;
if (x < nhub) {
int nid = hub[x];
float sum = 0.0;
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
neighbor_accum_weight_result[eid] = sum;
}
sum_weight_result[nid] = sum;
}
}