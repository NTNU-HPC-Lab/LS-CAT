#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result, int width, int height){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int nid = y * width + x; // thread_index is node id
if (x < width && y < height && mask[nid]){
double sum = 0.0;
int end = min(neighbor_start[nid+1], neighbor_start[nid]+HUB_THREASHOLD+1); //+1 because HUB_THREASHOLD is out degree
for (int eid = neighbor_start[nid]; eid < end; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
}
sum_weight_result[nid] = sum;
}
}