#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "next_move_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *rat_count = NULL;
hipMalloc(&rat_count, XSIZE*YSIZE);
int *healthy_rat_count = NULL;
hipMalloc(&healthy_rat_count, XSIZE*YSIZE);
int *exposed_rat_count = NULL;
hipMalloc(&exposed_rat_count, XSIZE*YSIZE);
int *infectious_rat_count = NULL;
hipMalloc(&infectious_rat_count, XSIZE*YSIZE);
double *node_weight = NULL;
hipMalloc(&node_weight, XSIZE*YSIZE);
double *sum_weight_result = NULL;
hipMalloc(&sum_weight_result, XSIZE*YSIZE);
int *neighbor = NULL;
hipMalloc(&neighbor, XSIZE*YSIZE);
int *neighbor_start = NULL;
hipMalloc(&neighbor_start, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
double batch_fraction = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
next_move_kernel<<<gridBlock,threadBlock>>>(rat_count,healthy_rat_count,exposed_rat_count,infectious_rat_count,node_weight,sum_weight_result,neighbor,neighbor_start,width,height,batch_fraction);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
next_move_kernel<<<gridBlock,threadBlock>>>(rat_count,healthy_rat_count,exposed_rat_count,infectious_rat_count,node_weight,sum_weight_result,neighbor,neighbor_start,width,height,batch_fraction);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
next_move_kernel<<<gridBlock,threadBlock>>>(rat_count,healthy_rat_count,exposed_rat_count,infectious_rat_count,node_weight,sum_weight_result,neighbor,neighbor_start,width,height,batch_fraction);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}