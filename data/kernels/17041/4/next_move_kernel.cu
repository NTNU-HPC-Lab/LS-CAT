#include "hip/hip_runtime.h"
#include "includes.h"




#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

#define TRANSITION_PROB 0.02 * 0.1

__global__ void next_move_kernel(int *rat_count, int *healthy_rat_count, int *exposed_rat_count, int *infectious_rat_count, double *node_weight, double *sum_weight_result,int *neighbor, int *neighbor_start,  int width, int height, double batch_fraction){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int nid = y * width + x;
if (x < width && y < height){
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) {
int remote_node = neighbor[eid];
double move_prob = batch_fraction * node_weight[remote_node] / sum_weight_result[nid]; // check 0
int move_rat = rat_count[nid] * move_prob;
int move_healthy = healthy_rat_count[nid] * move_prob;
int move_exposed = exposed_rat_count[nid] * move_prob;
int move_infectious = infectious_rat_count[nid] * move_prob;
atomicAdd(&rat_count[remote_node], move_rat);
atomicAdd(&healthy_rat_count[remote_node], move_healthy);
atomicAdd(&exposed_rat_count[remote_node], move_exposed);
atomicAdd(&infectious_rat_count[remote_node], move_infectious);
rat_count[nid] -= move_rat;
healthy_rat_count[nid] -= move_healthy;
exposed_rat_count[nid] -= move_exposed;
infectious_rat_count[nid] -= move_infectious;
}
}
}