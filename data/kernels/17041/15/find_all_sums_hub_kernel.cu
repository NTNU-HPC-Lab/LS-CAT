#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result){
int x = blockIdx.x * blockDim.x + threadIdx.x;
if (x < nhub) {
int nid = hub[x];
double sum = 0.0;
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
neighbor_accum_weight_result[eid] = sum;
}
sum_weight_result[nid] = sum;
}
}