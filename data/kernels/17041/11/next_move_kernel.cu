#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void next_move_kernel(bool *mask, int *rat_count, int *healthy_rat_count, int *exposed_rat_count, int *infectious_rat_count, double *node_weight, double *sum_weight_result,int *neighbor, int *neighbor_start,  int width, int height, double batch_fraction){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int nid = y * width + x;
if (x < width && y < height){
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid] + 5; eid++) { // 5 because self + up down left right
int remote_node = neighbor[eid];
double move_prob = batch_fraction * node_weight[remote_node] / sum_weight_result[nid]; // check 0
int move_rat = rat_count[nid] * move_prob;
int move_healthy = healthy_rat_count[nid] * move_prob;
int move_exposed = exposed_rat_count[nid] * move_prob;
int move_infectious = infectious_rat_count[nid] * move_prob;
atomicAdd(&rat_count[remote_node], move_rat);
atomicAdd(&healthy_rat_count[remote_node], move_healthy);
atomicAdd(&exposed_rat_count[remote_node], move_exposed);
atomicAdd(&infectious_rat_count[remote_node], move_infectious);
rat_count[nid] -= move_rat;
healthy_rat_count[nid] -= move_healthy;
exposed_rat_count[nid] -= move_exposed;
infectious_rat_count[nid] -= move_infectious;
}
}
}