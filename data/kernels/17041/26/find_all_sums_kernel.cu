#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_all_sums_kernel(bool *mask, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result, int width, int height){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int nid = y * width + x; // thread_index is node id
if (x < width && y < height && mask[nid]){
float sum = 0.0;
int end = min(neighbor_start[nid+1], neighbor_start[nid]+HUB_THREASHOLD+1); //+1 because HUB_THREASHOLD is out degree
for (int eid = neighbor_start[nid]; eid < end; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
neighbor_accum_weight_result[eid] = sum;
}
sum_weight_result[nid] = sum;
}
}