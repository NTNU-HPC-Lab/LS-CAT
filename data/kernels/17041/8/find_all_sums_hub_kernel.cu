#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result){
int x = blockIdx.x * blockDim.x + threadIdx.x;
if (x < nhub) {
int nid = hub[x];
double sum = 0.0;
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
}
sum_weight_result[nid] = sum;
}
}