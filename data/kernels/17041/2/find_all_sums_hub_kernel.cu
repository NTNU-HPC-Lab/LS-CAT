#include "hip/hip_runtime.h"
#include "includes.h"




#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

#define TRANSITION_PROB 0.02 * 0.1

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result){
int x = blockIdx.x * blockDim.x + threadIdx.x;
if (x < nhub) {
int nid = hub[x];
double sum = 0.0;
for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) { // this eid is just index of the neighbor in the neighbor array
sum += node_weight[neighbor[eid]];
}
sum_weight_result[nid] = sum;
}
}