#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_infection_prob_kernel(double alpha, double beta, int *infectious_rat_count, int *exposed_rat_count, int *rat_count, double *infection_prob_result, int width, int height) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int nid = y * width + x;
if(x < width && y < height) {
if(rat_count[nid] == 0) {
infection_prob_result[nid] = 0.0;
} else {
double density_of_exposed = (double)(exposed_rat_count[nid]) / (double)(rat_count[nid]);
double density_of_infectious = (double)(infectious_rat_count[nid]) / (double)(rat_count[nid]);
infection_prob_result[nid] = alpha * density_of_infectious + beta * density_of_exposed;
}
}
}