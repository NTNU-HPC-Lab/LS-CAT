#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
// insert operation here
deviceResult[i] = deviceA[i]+deviceB[i];
}