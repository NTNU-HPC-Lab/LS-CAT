#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void scan(int *v, const int n)
{
int tIdx = threadIdx.x;
int step = 1;

while (step < n) {

int indiceDroite = tIdx;
int indiceGauche = indiceDroite + step;

if (indiceGauche < n) {
v[indiceDroite] = v[indiceDroite] + v[indiceGauche];
}

step = step * 2;
__syncthreads();

}

}