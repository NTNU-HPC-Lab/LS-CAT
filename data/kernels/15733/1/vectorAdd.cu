#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(int* a, int* b, int* c, int n) {
// Calculate global thread ID (tid)
int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
// Vector boundary guard
if (tid < n) {
// Each thread adds a single element
c[tid] = a[tid] + b[tid];
}
}