#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_1d(int* a, int* c, int n) {
int id = blockIdx.x * blockDim.x + threadIdx.x;

//cal the radius of the mask(mid point)
int r = MASK_LEN / 2;
//cal the start point of for the element
int start = id - r;
int temp = 0;
for (int j = 0; j < MASK_LEN; j++)
{
if ((start + j >= 0) && (start + j < n))
{
temp += a[start + j] * mask[j];
}
}
c[id] = temp;
}