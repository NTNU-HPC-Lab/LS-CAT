#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_2d(int* Mat, int* res, int n) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int start_r = row - MASK_OFFSET;
int start_c = col - MASK_OFFSET;

int temp = 0;

for (int i = 0; i < MASK_LEN; i++)
{
for (int j = 0; j < MASK_LEN; j++)
{
if ((start_r + i >= 0) && (start_r + i < n))
{
if ((start_c + j >= 0) && (start_c + j < n))
{
temp += Mat[(start_r + i) * n + (start_c + j)] * mask[i * MASK_LEN + j];
}
}
}
}

res[row * n + col] = temp;
}