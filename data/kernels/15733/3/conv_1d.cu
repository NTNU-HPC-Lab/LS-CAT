#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_1d(int* a, int* b, int* c, int n, int m) {
int id = blockIdx.x * blockDim.x + threadIdx.x;

//cal the radius of the mask(mid point)
int r = m / 2;
//cal the start point of for the element
int start = id - r;
int temp = 0;
for (int j = 0; j < m; j++)
{
if ((start + j >= 0) && (start + j < n))
{
temp += a[start + j] * b[j];
}
}
c[id] = temp;
}