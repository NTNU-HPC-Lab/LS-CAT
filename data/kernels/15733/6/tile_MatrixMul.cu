#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tile_MatrixMul(int* a, int* b, int* c, int n, int tile_size) {
//statically-sized memory
__shared__ int A[Shared_Mem_Size];
__shared__ int B[Shared_Mem_Size];

int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x;
int by = blockIdx.y;

//cal global row and col postions for this thread
int row = by * tile_size + ty;
int col = bx * tile_size + tx;

//Intermidiate sum for element being written
int temp_val = 0;

//sweet tiles over entire matrix
for (int i = 0; i < (n / tile_size); i++)
{
/*

Every thread in a threadblock loads one element into shared memory
The element location in shared memory corresponds to the thread's
position in the threadblock (e.g thread[0,0] loads for
A[0 * tile_size + 0] and B[0 * tile_size + 0])

Explanation of indexing parameters
for A:
row*n: Indexes the global row for this thread (loop invariant)
i*tile_size: Indexes new set of column each iteration
tx: Indexes the column within that set

for B:
col: Indexes the global column this thread (loop invariant)
i*tile_size*n: Indexes next set of rows each iteration
ty*n: Indexes the row within that set
*/
A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col];

//Ensure all threads have loaded their data before proceeding
__syncthreads();

//cal all temp values for this tile
for (int j = 0; j < tile_size; j++)
{
temp_val += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
}

//Ensure some threads dont progress and stomp current shared memory values
__syncthreads();
}
c[(row * n) + col] = temp_val;
}