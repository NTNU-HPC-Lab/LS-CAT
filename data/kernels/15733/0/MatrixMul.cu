#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel for vector addition

// Initialize
__global__ void MatrixMul(int* a, int* b, int* c, int n) {
// row
int row = (blockIdx.y * blockDim.y) + threadIdx.y;
//col
int col = (blockIdx.x * blockDim.x) + threadIdx.x;
int temp_sum = 0;
// boundary guard
if ((row < n) && (col < n)) {
for (int k = 0; k < n; k++)
{
temp_sum += a[row*n+k]*b[k*n+col];
}
c[row*n+col] = temp_sum;
}
}