#include "hip/hip_runtime.h"
#include "includes.h"

#define getPos(a,k) (((a)>>(k-1))&1)

extern "C" {



}
__global__ void prefixSum(int * input_T, int * prefix_T, int * prefix_helper_T, int n, int k, int blockPower) {
__shared__ int tmp_T[1024];

for(int i = 0; i<blockPower; i++) {
if(threadIdx.x + 1024*blockIdx.x + i*1024*gridDim.x >= n) return;

tmp_T[threadIdx.x] = input_T[threadIdx.x + 1024*blockIdx.x + i*1024*gridDim.x];
tmp_T[threadIdx.x] = getPos(tmp_T[threadIdx.x],k);

int val,kk = 1;
while(kk <= 512) {
__syncthreads();
if(kk <= threadIdx.x) val = tmp_T[threadIdx.x - kk];
__syncthreads();
if(kk <= threadIdx.x) tmp_T[threadIdx.x] += val;
kk *= 2;
}

__syncthreads();

prefix_T[threadIdx.x + 1024*blockIdx.x + i*1024*gridDim.x] = tmp_T[threadIdx.x];

if(threadIdx.x == 1023 || threadIdx.x + 1024*blockIdx.x + i*1024*gridDim.x == n-1) prefix_helper_T[i*gridDim.x + blockIdx.x + 1] = tmp_T[threadIdx.x];
}
}