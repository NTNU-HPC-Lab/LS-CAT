#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 2147483647

extern "C" {





}
__global__ void oneMove(int * tab, int dist, int pow, int blocksPerTask, int period) {
__shared__ int tmp_T[1024];
__shared__ int begin;

if(threadIdx.x == 0)
begin = (blockIdx.x/blocksPerTask)*dist*2 + (blockIdx.x%blocksPerTask)*512*pow;

__syncthreads();

if((blockIdx.x / period) % 2 == 0) {
for(int i = begin; i < begin + pow*512; i += 512) {
if(threadIdx.x < 512) tmp_T[threadIdx.x] = tab[i + threadIdx.x];
else tmp_T[threadIdx.x] = tab[i + threadIdx.x - 512 + dist];

__syncthreads();

if(threadIdx.x < 512 && tmp_T[threadIdx.x] > tmp_T[threadIdx.x + 512]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
tmp_T[threadIdx.x + 512] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
}

__syncthreads();

if(threadIdx.x < 512) tab[i + threadIdx.x] = tmp_T[threadIdx.x];
else tab[i + threadIdx.x - 512 + dist] = tmp_T[threadIdx.x];

__syncthreads();
}
} else {
for(int i = begin; i < begin + pow*512; i += 512) {
if(threadIdx.x < 512) tmp_T[threadIdx.x] = tab[i + threadIdx.x];
else tmp_T[threadIdx.x] = tab[i + threadIdx.x - 512 + dist];

__syncthreads();

if(threadIdx.x < 512 && tmp_T[threadIdx.x] < tmp_T[threadIdx.x + 512]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
tmp_T[threadIdx.x + 512] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
}

__syncthreads();

if(threadIdx.x < 512) tab[i + threadIdx.x] = tmp_T[threadIdx.x];
else tab[i + threadIdx.x - 512 + dist] = tmp_T[threadIdx.x];

__syncthreads();
}
}
}