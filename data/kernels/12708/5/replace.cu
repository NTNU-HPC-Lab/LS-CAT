#include "hip/hip_runtime.h"
#include "includes.h"

#define getPos(a,k) (((a)>>(k-1))&1)

extern "C" {



}
__global__ void replace(int * input_T, int * output_T, int * prefix_T, int * prefix_helper_T, int n, int k, int blockPower) {
for(int i = 0; i<blockPower; i++) {
int oldpos = threadIdx.x + 1024*blockIdx.x + i*1024*gridDim.x;
if(oldpos >= n) return ;


int newpos = prefix_T[oldpos] + prefix_helper_T[blockIdx.x + i*gridDim.x];

if(getPos(input_T[oldpos],k) == 0) {
newpos = oldpos - newpos;
} else {
newpos = prefix_helper_T[(n+1023)/1024] + newpos - 1;
}

output_T[newpos] = input_T[oldpos];
}

}