#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 2147483647

extern "C" {





}
__global__ void oneReduction(int * tab, int len, int mod) {

__shared__ int begin, end;
__shared__ int tmp_T[1024];

if(threadIdx.x == 0) {
begin = blockIdx.x*len;
end = blockIdx.x*len + len;
}

__syncthreads();

if(blockIdx.x % mod < mod/2) {
for(int k = len/2; k >= 1024; k /= 2) {
for(int g = begin; g < end; g += 2*k) {
for(int j = g; j < g + k; j += 512) {
__syncthreads();

if(threadIdx.x < 512)
tmp_T[threadIdx.x] = tab[j + threadIdx.x];
else
tmp_T[threadIdx.x] = tab[j + threadIdx.x - 512 + k];

__syncthreads();
if(threadIdx.x < 512 && tmp_T[threadIdx.x] > tmp_T[threadIdx.x + 512]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
tmp_T[threadIdx.x + 512] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
}

__syncthreads();
if(threadIdx.x < 512)
tab[j + threadIdx.x] = tmp_T[threadIdx.x];
else
tab[j + threadIdx.x - 512 + k] = tmp_T[threadIdx.x];
}
}
}

for(int i = begin; i < begin+len; i += 1024) {
__syncthreads();
tmp_T[threadIdx.x] = tab[i + threadIdx.x];
__syncthreads();
for(int jump = 512; jump >= 1; jump /= 2) {
if(threadIdx.x % (jump*2) < jump && threadIdx.x + jump < 1024  && tmp_T[threadIdx.x] > tmp_T[threadIdx.x + jump]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + jump];
tmp_T[threadIdx.x + jump] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + jump];
}
__syncthreads();
}
tab[i + threadIdx.x] = tmp_T[threadIdx.x];
}
} else {
for(int k = len/2; k >= 1024; k /= 2) {
for(int g = begin; g < end; g += 2*k) {
for(int j = g; j < g + k; j += 512) {
__syncthreads();
if(threadIdx.x < 512)
tmp_T[threadIdx.x] = tab[j + threadIdx.x];
else
tmp_T[threadIdx.x] = tab[j + threadIdx.x - 512 + k];

__syncthreads();
if(threadIdx.x < 512 && tmp_T[threadIdx.x] < tmp_T[threadIdx.x + 512]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
tmp_T[threadIdx.x + 512] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + 512];
}

__syncthreads();
if(threadIdx.x < 512)
tab[j + threadIdx.x] = tmp_T[threadIdx.x];
else
tab[j + threadIdx.x - 512 + k] = tmp_T[threadIdx.x];
}
}
}

for(int i = begin; i < begin + len; i += 1024) {
__syncthreads();
tmp_T[threadIdx.x] = tab[i + threadIdx.x];
__syncthreads();
for(int jump = 512; jump >= 1; jump /= 2) {
if(threadIdx.x % (jump*2) < jump && threadIdx.x + jump < 1024  && tmp_T[threadIdx.x] < tmp_T[threadIdx.x + jump]) {
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + jump];
tmp_T[threadIdx.x + jump] ^= tmp_T[threadIdx.x];
tmp_T[threadIdx.x] ^= tmp_T[threadIdx.x + jump];
}
__syncthreads();
}
tab[i + threadIdx.x] = tmp_T[threadIdx.x];
}
}


}