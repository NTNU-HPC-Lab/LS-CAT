#include "hip/hip_runtime.h"
#include "includes.h"
//=============================================================================
// FILE:   mytoy.cu
// AUTHORS: Raul Segura & Manuel Ujaldon (copyright 2014)
// Look for the string "MU" whenever Manuel suggests you to introduce changes
// Feel free to change some other parts of the code too (at your own risk)
//=============================================================================



//=============================================================================
// CUDA functions.
//=============================================================================

//Error handler for CUDA functions.
__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

// "numOperations" is the 2nd input parameter to our executable
if (vi < nextColInd) {
for (int j=0; j<numOperations; ++j) {
// The operation performed on each nonzero of our sparse matrix:
dvalues[vi] *=dvalues[vi]+dvalues[vi]*dvalues[vi]; // POINT 3: Choices you may try here:
}                               // *= (for multiply), /= (for division),
}                                   // or you may investigate some other :-)
}