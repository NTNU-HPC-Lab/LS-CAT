#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscaleKernel(int *ms, int *aux, int n){
int i = threadIdx.x+blockDim.x*blockIdx.x;
int k=0;

int grayscale=0;
if(i<n){
for(k=0; k<n-3; k+=3){
grayscale = 0.299*ms[i*n+k] + 0.5876*ms[i*n+k+1] + 0.114*ms[i*n+k+2];
aux[i*n+k] = aux[i*n+k+1] = aux[i*n+k+2] = grayscale;
}
}
}