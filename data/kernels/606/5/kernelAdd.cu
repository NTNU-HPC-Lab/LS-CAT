#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

// "numOperations" is the 2nd input parameter to our executable
if (vi < nextColInd) {
for (int j=0; j<numOperations; ++j) {
// The operation performed on each nonzero of our sparse matrix:
dvalues[vi] /=dvalues[vi]+dvalues[vi]*dvalues[vi]; // POINT 3: Choices you may try here:
}                               // *= (for multiply), /= (for division),
}                                   // or you may investigate some other :-)
}