#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void twiddleImgKernelIDFT(float *wi, float *w, int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int i, index;

if (idx < N) {
if (idx == 0) {
for (i = 0; i < N; i++)
wi[idx * N + i] = 0;
} else {
wi[idx * N + 0] = 0;
for (i = 1; i < N; i++) {
index = (idx * i) % N;
wi[idx * N + i] = w[index * 2 + 1];
}
}
}
}