#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inputKernel(float *x, int N)
{
int ix   = blockIdx.x * blockDim.x + threadIdx.x;
int iy   = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * NUM_OF_X_THREADS + ix;

if (idx < N)
x[idx]  = x[idx] + (float)idx;
}