#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divMat(float *a, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if((idx*N) < (N*N))
a[idx *N] /= N;
}