#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dotProdKernel(float *a, float *b, float *ab, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if( (idx*N) < (N*N) ) {
ab[idx * N] = a[idx *N] * b[idx * N];
}
}