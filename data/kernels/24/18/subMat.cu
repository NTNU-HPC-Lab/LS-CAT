#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subMat(float *a, float *b, float *sub, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if((idx*N) < (N*N))
sub[idx * N] = a[idx * N] - b[idx * N];
}