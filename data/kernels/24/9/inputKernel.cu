#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inputKernel(float *x, int n, int N)
{
int ix   = blockIdx.x * blockDim.x + threadIdx.x,i;
int iy   = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * NUM_OF_X_THREADS + ix;

if (idx < N)
{
if (idx < n)
{
x[idx*N]  = (float)idx;
}
else
{
x[idx] = 0;
}

for(i=1;i<N;i++)
{
x[idx*N + i] = 0;
}
}

}