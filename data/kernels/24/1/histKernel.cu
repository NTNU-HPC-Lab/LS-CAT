#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histKernel(char *inData, long size, unsigned int *histo)
{
__shared__ unsigned int temp[BIN_COUNT][BIN_COUNT];
__shared__ unsigned int blockSum[BIN_COUNT];
int i = 0;

while(i < BIN_COUNT)
temp[i++][threadIdx.x] = 0;

__syncthreads();

int tid = threadIdx.x + blockIdx.x * blockDim.x;
int offset = blockDim.x * gridDim.x;

while(tid < size) {
temp[(int)inData[tid]][threadIdx.x]++;
tid += offset;
}

__syncthreads();

i = 0;
while(i < BIN_COUNT)
blockSum[threadIdx.x] += temp[threadIdx.x][i++];

__syncthreads();

atomicAdd(&(histo[threadIdx.x]), blockSum[threadIdx.x]);
}