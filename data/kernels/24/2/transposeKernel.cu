#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeKernel(float *inData, float *outData)
{
__shared__ float tile[TILE_DIM][TILE_DIM + 1];

int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

/* Copying data into shared memory - each thread copies 4 elements : read & write coalesced */
for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
tile[threadIdx.y + j][threadIdx.x] = inData[(y+j) * width + x];

__syncthreads();

/* x,y modified according to the new transposed matrix */
x = blockIdx.y * TILE_DIM + threadIdx.x;
y = blockIdx.x * TILE_DIM + threadIdx.y;

/* Copying data to output array - each thread copies 4 elemets : read & write coalesced */
for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
outData[(y+j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}