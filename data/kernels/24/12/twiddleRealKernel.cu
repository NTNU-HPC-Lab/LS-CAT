#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void twiddleRealKernel(float *wr, float *w, int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int i = 0, index;

if (idx < N) {
if (idx == 0) {
for (i = 0; i < N; i++)
wr[idx * N + i] = 1;
} else {
wr[idx * N + 0] = 1;
for (i = 1; i < N; i++) {
index = (idx * i) % N;
wr[idx * N + i] = w[index * 2];
}
}
}
}