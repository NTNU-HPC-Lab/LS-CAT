#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMat(float *a, float *b, float *add, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if((idx*N) < (N*N))
add[idx * N] = a[idx *N] + b[idx * N];
}