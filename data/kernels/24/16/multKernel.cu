#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multKernel(float *a, float *b, float *ab, int width)
{
int tx = threadIdx.x, ty = threadIdx.y;
int bx = blockIdx.x, by = blockIdx.y;

// allocate tiles in __shared__ memory
__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

// calculate the row & col index to identify element to work on
int row = by*blockDim.y + ty;
int col = bx*blockDim.x + tx;
float result = 0;

// loop over the tiles of the input in phases
for(int p = 0; p < width/TILE_WIDTH; ++p)
{
// collaboratively load tiles into shared memory: row-wise and column wise respectively
s_a[ty][tx] = a[row*width + (p*TILE_WIDTH + tx)];
s_b[ty][tx] = b[(p*TILE_WIDTH + ty)*width + col];
__syncthreads();

// dot product between row of s_a and col of s_b
for(int k = 0; k < TILE_WIDTH; ++k)
result += s_a[ty][k] * s_b[k][tx];
__syncthreads();
}
ab[row*width+col] = result;
}