#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void factorKernel(float *w, int N)
{
int ix  = blockIdx.x * blockDim.x + threadIdx.x;
int idx = ix * 2;
int izx = N + idx;

const float pi = 3.141592653589793238462643383;
float aw = (2.0 * pi) / (float)N;
float arg = aw * (float)ix;

/* Twiddle factors are symmetric along N/2. with change in sign, due to 180 degree phase change */
if (idx < N) {
w[idx] = cos(arg);
w[idx + 1] = sin(arg);
w[izx] = (-1) * w[idx];
w[izx+1] = (-1) * w[idx + 1];
}
}