#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward1(float *zs,int bound2,float *weights,int w_off,float *activations1) {

int i;

zs[threadIdx.x]=0.0;
for (i=0; i<bound2; i++) {
zs[threadIdx.x]+=weights[w_off+(threadIdx.x*bound2)+i]*activations1[i];
}
}