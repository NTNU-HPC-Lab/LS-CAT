#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube(float * d_out, float * d_in){
int id = threadIdx.x;
float num = d_in[id];
d_out[id] = num * num * num;
}