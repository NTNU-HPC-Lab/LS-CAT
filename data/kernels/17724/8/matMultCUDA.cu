#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
__shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
const int tidc = threadIdx.x;
const int tidr = threadIdx.y;
const int bidc = blockIdx.x*BLOCK_SIZE;
const int bidr = blockIdx.y*BLOCK_SIZE;
int i,j;
float results = 0;
float comp = 0;
for (j=0; j<n; j+=BLOCK_SIZE) {
matA[tidr][tidc] = a[(tidr+bidr)*lda+tidc+j];
matB[tidr][tidc] = b[(tidr+j)*ldb+tidc+bidc];

__syncthreads();

for (i=0;i<BLOCK_SIZE;i++){
float t;
comp -= matA[tidr][i]*matB[i][tidc];
t=results-comp;
comp=(t-results)+comp;
results=t;
}

__syncthreads();
}
c[(tidr+bidr)*ldc+tidc+bidc]=results;
}