#include "hip/hip_runtime.h"
#include "includes.h"


// Device input vectors
int *d_a;
//Device output vector
int *d_b;








__global__ void setLastToCero(int *A, int size) {
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index == size - 1) {
A[index] = 0;
}
}