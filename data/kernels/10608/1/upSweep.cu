#include "hip/hip_runtime.h"
#include "includes.h"


// Device input vectors
int *d_a;
//Device output vector
int *d_b;








__global__ void upSweep(int *A, int size, int iteration) {
const int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
if (!((index + 1) % (1 << (iteration + 1))))
A[index] = A[index - (1<<iteration)] + A[index];
}

}