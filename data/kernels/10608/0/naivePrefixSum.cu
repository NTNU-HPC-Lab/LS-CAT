#include "hip/hip_runtime.h"
#include "includes.h"


// Device input vectors
int *d_a;
//Device output vector
int *d_b;








__global__ void naivePrefixSum(int *A, int *B, int size, int iteration) {
const int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
if (index >= (1 << (iteration - 1)))
A[index] = B[(int) (index - (1 << (iteration - 1)))] + B[index];
else
A[index] = B[index];

}
}