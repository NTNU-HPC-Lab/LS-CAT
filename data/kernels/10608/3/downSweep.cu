#include "hip/hip_runtime.h"
#include "includes.h"


// Device input vectors
int *d_a;
//Device output vector
int *d_b;








__global__ void downSweep(int *A, int size, int iteration) {
const int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size) {
int aux;
if (!((index + 1) % (1 << (iteration + 1)))) {
aux = A[index - (1<<iteration)];
A[index - (1<<iteration)] = A[index];
A[index] = aux + A[index];
}
}
}