#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_mul_shared(float *ad,float *bd,float *cd,int N)
{
float pvalue=0;
int TILE=blockDim.x;
int ty=threadIdx.y;
int tx=threadIdx.x;

//allocate shared memory per block
__shared__ float ads[16][16];
__shared__ float bds[16][16];

//find Row and Column corresponding to a data element for each thread
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

//iterate through TILEs to traverse whole WIDTH
for(int i=0;i< N/TILE;++i)
{
//copy values of data TILE into shared memory
ads[ty][tx] = ad[Row * N + (i * TILE) + tx];
bds[ty][tx] = bd[(i * TILE + ty) * N + Col];

__syncthreads();                            //synchronize to confirm that whole TILE has been copied

//calculate partial dot-product
for(int k=0;k<TILE;k++)
pvalue += ads[ty][k] * bds[k][tx];

__syncthreads();                            //synchronize to confirm that whole partial product corresponding to all threads of the block has been calculated
}

//store dot product at corresponding positon in resultant Matrix
cd[Row * N + Col] = pvalue;
}