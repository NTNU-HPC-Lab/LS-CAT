#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_mul(float *ad,float *bd,float *cd,int N)
{
float pvalue=0;

//find Row and Column corresponding to a data element for each thread
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

//calculate dot product of Row of First Matrix and Column of Second Matrix
for(int i=0;i< N;++i)
{
float m=ad[Row * N+i];
float n=bd[i * N + Col];
pvalue += m * n;
}

//store dot product at corresponding positon in resultant Matrix
cd[Row * N + Col] = pvalue;

}