#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_matrices(float *ad,float *bd,float *cd,int N)
{
cd[threadIdx.y * N + threadIdx.x] = ad[threadIdx.y * N + threadIdx.x] + bd[threadIdx.y * N + threadIdx.x];
}