#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *ad, int N)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < N)
ad[index] *= ad[index];                             //adding values in GPU memory
}