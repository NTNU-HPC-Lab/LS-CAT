#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_vectors(float *ad, float *bd, int N)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < N)
ad[index] += bd[index];                             //adding values in GPU memory
}