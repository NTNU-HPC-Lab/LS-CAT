#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void my_memsetIdx(int* dg_array, int size, int scale){
const int gtid=blockIdx.x*blockDim.x + threadIdx.x;
if(gtid < size){
dg_array[gtid] = gtid*scale;
}
}