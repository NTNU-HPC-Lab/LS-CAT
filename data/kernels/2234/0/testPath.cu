#include "hip/hip_runtime.h"
#include "includes.h"
#define MAX_STEPS 32


void readFile();
int cpuPathTest(int limitSteps, unsigned long long *tid);
void printMaze();
void printPath(unsigned long long tid, int steps);
void printPathMaze(unsigned long long tid, int steps);
void setTime0();
void getExeTime();

struct Maze
{
char maze[99][99];
int rows, cols, s_x, s_y, e_x, e_y;
};

struct Maze maze;
FILE *MAZE;
struct timespec t_start, t_end;
double elapsedTime;

const int threadsPerBlock = 1024;
const int blocksPerGrid = 1024;



__global__ void testPath(int *limitSteps, struct Maze *maze, int *workDone , unsigned long long *path)
{
unsigned long long tid = threadIdx.x + blockIdx.x * blockDim.x;
unsigned long long bias = blockDim.x * gridDim.x;
unsigned long long maxRoute = 0xffffffffffffffff - (bias - 1);//max length of path
maxRoute >>= (MAX_STEPS - *limitSteps) * 2;//(32 - 1) * 2 = 62 =>0~011 only 3 steps : right up left

while(tid <= maxRoute)
{
if(*workDone) break;

int x = maze->s_x, y = maze->s_y;
unsigned long long temp = tid;

int i = *limitSteps;
int steps = 0;
do
{
//GetMoveDirection
steps++;
int direction = temp & 3;//mask
temp >>= 2;
//Move
switch(direction)
{
case 0 :
x += 1;
break;
case 1 :
y -= 1;
break;
case 2 :
x -= 1;
break;
case 3 :
y += 1;
break;
}
//if at Target, print path ,else keep going, if no way then break
if(maze->maze[y][x] == '$')
{
*workDone = 1;
*path = tid;
break;
}
else if(maze->maze[y][x] != '.')
{
break;
}
}
while(i--);

tid += bias;
}
}