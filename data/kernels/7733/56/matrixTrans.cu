#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTrans(float * M,float * MT)
{
int val=0;

int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

MT[row + col*N] = 0;
if (row < N && col < N)
{
val = M[col + row*N];
MT[row + col*N] = val;

}
}