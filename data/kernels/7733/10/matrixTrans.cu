#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTrans(double * M,double * MT, int rows, int cols)
{
double val=0;
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if (row < rows && col < cols){
val = M[col + row*cols];
MT[row + col*rows] = val;
}
}