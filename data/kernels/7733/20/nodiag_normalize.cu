#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nodiag_normalize(double *A, double *I, int nn, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if ( x< nn && y < nn){
if (x < nn && y < nn){
if (x == i && x!=y){
I[x*nn + y] /= A[i*nn + i];
A[x*nn + y] /= A[i*nn + i];
}
}
}
}