#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_zero(float *A, float *I, int n, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n){
if (x != i){
if (y == i){
A[x*n + y] = 0;
}
}
}
}