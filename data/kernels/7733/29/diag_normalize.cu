#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diag_normalize(double *A, double *I, int nn, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < nn && y < nn){
if (x == y && x == i){
I[x*nn + y] /= A[i*nn + i];
A[x*nn + y] /= A[i*nn + i];
}
}
}