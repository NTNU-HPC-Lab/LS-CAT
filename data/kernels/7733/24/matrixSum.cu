#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSum(const double * M1,const double * M2,double * Msum,double alpha,double beta, int rows, int cols)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;
if (row < rows && col < cols){
Msum[row + col*rows] = alpha*M1[row+col*rows]+beta*M2[row+col*rows];
}
}