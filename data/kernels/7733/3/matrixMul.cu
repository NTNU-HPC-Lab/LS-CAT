#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192


//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void matrixMul(double * a,double * b, double * C, int cols,int rows,int cols2)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;
if (row < rows && col < cols)
{
C[row*cols+col]  =0;
for (int k = 0; k < cols2; k++)
{
C[row*cols+col]+=b[k*cols+col]*a[row*cols2+k];
}
}
}