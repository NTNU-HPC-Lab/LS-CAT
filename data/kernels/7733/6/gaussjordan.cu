#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192


//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if ( x< nn && y < nn)
{
if (x < nn && y < nn)
{
if (x != i)
{
I[x*nn + y] -= I[i*nn + y] * A[x*nn + i];
if (y != i)
{
A[x*nn + y] -= A[i*nn + y] * A[x*nn + i];
}
}
}
}
}