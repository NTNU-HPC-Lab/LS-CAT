#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nodiag_normalize(float *A, float *I, int n, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n)
if (x == i && x!=y){
I[x*n + y] /= A[i*n + i];
A[x*n + y] /= A[i*n + i];
}

}