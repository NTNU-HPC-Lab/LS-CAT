#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(double * a,double * b, double * C, int cols,int rows,int cols2)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if (row < rows && col < cols){
for (int k = 0; k < cols2; k++){
C[row*cols+col]+=b[k*cols+col]*a[row*cols2+k];
}
}
}