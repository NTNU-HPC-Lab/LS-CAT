#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192


//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void matrixTrans(double * M,double * MT, int rows, int cols)
{
double val=0;
int col = blockIdx.y * blockDim.y + threadIdx.y;
int row = blockIdx.x * blockDim.x + threadIdx.x;

if (row < rows && col < cols)
{
val = M[col + row*cols];
MT[row + col*rows] = val;
}
}