#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192


//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void NOVA(double * Beta,double * Inverse,int * Vec, int p0,double Sigma2)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
double t0,Pvalue;
t0=Beta[x]/sqrt(Sigma2*Inverse[x*p0+x]);
Pvalue=2.*(1.-erf(t0));
if(Pvalue<0.25)
{
Vec[x]=1;
}
else
{
Vec[x]=0;
}
}