#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if ( x< nn && y < nn){

if (x < nn && y < nn){
if (x != i){
I[x*nn + y] -= I[i*nn + y] * A[x*nn + i];
if (y != i){
A[x*nn + y] -= A[i*nn + y] * A[x*nn + i];
}
}
}
}
}