#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kWhere(float* condition_mat, float* if_mat, float* else_mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
target[i] = condition_mat[i] ? if_mat[i] : else_mat[i];
}
}