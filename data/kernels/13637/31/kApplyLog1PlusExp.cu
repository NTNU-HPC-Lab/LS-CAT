#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
float mat_i;

for (unsigned int i = idx; i < len; i += numThreads) {
mat_i = mat[i];
if (mat_i > 0)
target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
else
target[i] = __logf(1 + __expf(mat_i));
}
}