#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSparseDot(int m, int n, int k, float *data, int* indptr, int* indices, float *dense_data, float* target, float beta, float alpha) {

const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
if (row < m && col < n) {
const int start = indptr[row];
const int end = indptr[row + 1];
float sum = 0;
for (int i = start; i < end; i++) {
sum += data[i]  * dense_data[col * k + indices[i]];
}
const int pos = col * m + row;
target[pos] = alpha * sum + ((beta == 0) ? 0 : beta * target[pos]);
}
}