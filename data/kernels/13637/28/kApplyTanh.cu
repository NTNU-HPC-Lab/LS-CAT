#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
float mat_i, exp2x;

for (unsigned int i = idx; i < len; i += numThreads) {
mat_i = mat[i];
exp2x = __expf(2 * mat_i);
target[i] = 1 - 2 / (exp2x + 1);
}
}