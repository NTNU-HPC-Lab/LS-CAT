#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateEigenVector(float* d_b, float* d_temp, float* normAb, int n)
{
int index = threadIdx.x + blockDim.x * blockIdx.x;
int stride = 0;

while (index + stride < n) {
d_b[index] = d_temp[index] / *normAb;

stride += blockDim.x * gridDim.x;
}
}