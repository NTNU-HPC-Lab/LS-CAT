#include "hip/hip_runtime.h"
#include "includes.h"
// C++ 17 Includes:

// Project Includes:

// Defines:
__global__ void sum_dynamic_kernel(const int* pIn, int* pOut, size_t numInts)
{
extern __shared__ int ps[]; // Automatically points to our shared memory array

// Load shared memory:
ps[threadIdx.x] = pIn[threadIdx.x];
if (threadIdx.x + blockDim.x <  numInts)
ps[threadIdx.x + blockDim.x] = pIn[threadIdx.x + blockDim.x];
if (0 == threadIdx.x && 1 == (1 & numInts))
ps[numInts - 1] = pIn[numInts - 1];

size_t prevNumThreads{numInts};
for (size_t numThreads{blockDim.x}; numThreads > 0; numThreads >>= 1)
{
if (threadIdx.x > numThreads)
return;

__syncthreads();
ps[threadIdx.x] += ps[threadIdx.x + numThreads];
if (1 == (prevNumThreads & 1))
ps[0] += ps[prevNumThreads - 1];

prevNumThreads = numThreads;
}

*pOut = ps[0];
}