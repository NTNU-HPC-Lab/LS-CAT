#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decrement_dynamic_kernel(int* pInts, size_t numInts)
{
size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx > numInts)
return;

pInts[idx] -= 1;
}