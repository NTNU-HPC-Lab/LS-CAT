#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Inc1(float *Ad, float *Bd){
// CHECK: int tx = hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x;
int tx = threadIdx.x + blockIdx.x * blockDim.x;
if(tx < 1 ){
for(int i=0;i<ITER;i++){
Ad[tx] = Ad[tx] + 1.0f;
for(int j=0;j<256;j++){
Bd[tx] = Ad[tx];
}
}
}
}