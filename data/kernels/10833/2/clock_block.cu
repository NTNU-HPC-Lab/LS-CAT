#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clock_block(clock_t *d_o, clock_t clock_count)
{
unsigned int start_clock = (unsigned int) clock();

clock_t clock_offset = 0;

while (clock_offset < clock_count)
{
unsigned int end_clock = (unsigned int) clock();

// The code below should work like
// this (thanks to modular arithmetics):
//
// clock_offset = (clock_t) (end_clock > start_clock ?
//                           end_clock - start_clock :
//                           end_clock + (0xffffffffu - start_clock));
//
// Indeed, let m = 2^32 then
// end - start = end + m - start (mod m).

clock_offset = (clock_t)(end_clock - start_clock);
}

d_o[0] = clock_offset;
}