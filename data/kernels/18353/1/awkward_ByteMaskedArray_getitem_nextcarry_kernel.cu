#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void awkward_ByteMaskedArray_getitem_nextcarry_kernel(int64_t* prefixed_mask, int64_t* to_carry, int8_t* mask, int64_t length) {
int64_t block_id =
blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int64_t thread_id = block_id * blockDim.x + threadIdx.x;

if(thread_id < length) {
if (mask[thread_id] != 0) {
to_carry[prefixed_mask[thread_id] - 1] = thread_id;
}
}
}