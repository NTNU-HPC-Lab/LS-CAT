#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void awkward_ByteMaskedArray_getitem_nextcarry_filter_mask(int8_t* mask, bool validwhen, int64_t length) {
int64_t block_id =
blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int64_t thread_id = block_id * blockDim.x + threadIdx.x;

if(thread_id < length) {
if ((mask[thread_id] != 0) == validwhen) {
mask[thread_id] = 1;
}
}
}