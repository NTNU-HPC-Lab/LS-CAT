#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void awkward_Content_getitem_next_missing_jagged_getmaskstartstop_kernel( int64_t* prefixed_index, int64_t* index_in, int64_t* offsets_in, int64_t* mask_out, int64_t* starts_out, int64_t* stops_out, int64_t length) {
int64_t block_id =
blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int64_t thread_id = block_id * blockDim.x + threadIdx.x;

if(thread_id < length) {
int64_t pre_in = prefixed_index[thread_id] - 1;
starts_out[thread_id] = offsets_in[pre_in];

if (index_in[thread_id] < 0) {
mask_out[thread_id] = -1;
stops_out[thread_id] = offsets_in[pre_in];
} else {
mask_out[thread_id] = thread_id;
stops_out[thread_id] = offsets_in[pre_in + 1];
}
}
}