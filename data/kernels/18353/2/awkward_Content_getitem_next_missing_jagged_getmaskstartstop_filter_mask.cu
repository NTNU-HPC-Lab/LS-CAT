#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void awkward_Content_getitem_next_missing_jagged_getmaskstartstop_filter_mask( int64_t* index_in, int64_t* filtered_index, int64_t length) {
int64_t block_id =
blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int64_t thread_id = block_id * blockDim.x + threadIdx.x;

if(thread_id < length) {
if (index_in[thread_id] >= 0) {
filtered_index[thread_id] = 1;
}
}
}