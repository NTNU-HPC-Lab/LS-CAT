#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyOffset(float *A, float *B, float *C, const int N, int offset)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i + offset] = A[i + offset] + B[i + offset];
}