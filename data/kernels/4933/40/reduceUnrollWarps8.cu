#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8
if (idx + 7 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
int b1 = g_idata[idx + 4 * blockDim.x];
int b2 = g_idata[idx + 5 * blockDim.x];
int b3 = g_idata[idx + 6 * blockDim.x];
int b4 = g_idata[idx + 7 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// unrolling warp
if (tid < 32)
{
volatile int *vmem = idata;
vmem[tid] += vmem[tid + 32];
vmem[tid] += vmem[tid + 16];
vmem[tid] += vmem[tid +  8];
vmem[tid] += vmem[tid +  4];
vmem[tid] += vmem[tid +  2];
vmem[tid] += vmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}