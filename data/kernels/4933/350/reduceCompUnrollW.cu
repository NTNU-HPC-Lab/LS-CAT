#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompUnrollW(int *g_idata, int *g_odata, unsigned int n)
{
// set the thread id.
unsigned int tid = threadIdx.x;
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;

// convert global data pointer to the local pointer of this block.
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8 data blocks.
if (idx + blockDim.x * 7 < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + blockDim.x * 2];
int a4 = g_idata[idx + blockDim.x * 3];
int b1 = g_idata[idx + blockDim.x * 4];
int b2 = g_idata[idx + blockDim.x * 5];
int b3 = g_idata[idx + blockDim.x * 6];
int b4 = g_idata[idx + blockDim.x * 7];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}
__syncthreads();

// in-place reduction and complete unroll
if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
__syncthreads();
if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
__syncthreads();
if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
__syncthreads();
if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vmem = idata;
vmem[tid] += vmem[tid + 32];
vmem[tid] += vmem[tid + 16];
vmem[tid] += vmem[tid +  8];
vmem[tid] += vmem[tid +  4];
vmem[tid] += vmem[tid +  2];
vmem[tid] += vmem[tid +  1];
}

// write result for this block to global mem.
if (tid == 0)
{
g_odata[blockIdx.x] = idata[0];
}
}