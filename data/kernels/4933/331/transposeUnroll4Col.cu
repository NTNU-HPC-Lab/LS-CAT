#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Col(int *in, int *out, const int nx, const int ny)
{
// set thread id.
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x * 4;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

unsigned int ti = iy * nx + ix; // access in rows.
unsigned int to = ix * ny + iy; // access in cols.

if (ix + 3 * blockDim.x < nx && iy < ny)
{
out[ti]                  = in[to];
out[ti + blockDim.x]     = in[to + ny * blockDim.x];
out[ti + blockDim.x * 2] = in[to + ny * blockDim.x * 2];
out[ti + blockDim.x * 3] = in[to + ny * blockDim.x * 3];
}
}