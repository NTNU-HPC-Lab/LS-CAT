#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
// set thread id.
unsigned int tid = threadIdx.x;
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

// convert global data pointer to th local pointer of this block.
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check.
if (idx >= n) return;

// in-place reduction in global memory.
for (int stride = 1; stride < blockDim.x; stride *= 2)
{
if ( (tid % (2 * stride)) == 0)
{
idata[tid] += idata[tid + stride];
}

// synchronize within block.
__syncthreads();
}

// write result for this block to global mem.
if (tid == 0)
{
g_odata[blockIdx.x] = idata[0];
}
}