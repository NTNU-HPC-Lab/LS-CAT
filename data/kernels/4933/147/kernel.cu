#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *F, double *D)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid == 0)
{
*F = 12.1;
*D = 12.1;
}
}