#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
// set the thread id.
unsigned int tid = threadIdx.x;
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

// convert global data pointer to the local pointer of this block.
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check.
if (idx >= n) return;

for (int stride = 1; stride < blockDim.x; stride *= 2)
{
// convert tid into local array index.
int index = 2 * stride * tid;

if (index < blockDim.x)
{
idata[index] += idata[index + stride];
}

// synchronize within threadblock.
__syncthreads();
}

// write result for this block to global mem.
if (tid == 0)
{
g_odata[blockIdx.x] = idata[0];
}
}