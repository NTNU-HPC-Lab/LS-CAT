#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned int n)
{
// set the thread id.
unsigned int tid = threadIdx.x;
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;

// convert global data pointer to the local pointer of this block.
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8 data blocks.
if (idx + blockDim.x * 7 < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + blockDim.x * 2];
int a4 = g_idata[idx + blockDim.x * 3];
int b1 = g_idata[idx + blockDim.x * 4];
int b2 = g_idata[idx + blockDim.x * 5];
int b3 = g_idata[idx + blockDim.x * 6];
int b4 = g_idata[idx + blockDim.x * 7];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}
__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x/2; stride>32; stride>>=1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock.
__syncthreads();
}

// unrolling warp
if (tid < 32)
{
volatile int *vmem = idata;
vmem[tid] += vmem[tid + 32];
vmem[tid] += vmem[tid + 16];
vmem[tid] += vmem[tid +  8];
vmem[tid] += vmem[tid +  4];
vmem[tid] += vmem[tid +  2];
vmem[tid] += vmem[tid +  1];
}

// write result for this block to global mem.
if (tid == 0)
{
g_odata[blockIdx.x] = idata[0];
}
}