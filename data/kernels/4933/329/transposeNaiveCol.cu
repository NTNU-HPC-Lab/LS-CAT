#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveCol(int *in, int *out, const int nx, const int ny)
{
// set thread id.
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < nx && iy < ny)
{
out[iy * nx + ix] = in[ix * ny + iy];
}
}