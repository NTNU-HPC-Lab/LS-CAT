#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
extern __shared__ int smem[];

// set thread ID
unsigned int tid = threadIdx.x;
int *idata = g_idata + blockIdx.x * blockDim.x;

// set to smem by each threads
smem[tid] = idata[tid];
__syncthreads();

// in-place reduction in global memory
if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vsmem = smem;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = smem[0];
}