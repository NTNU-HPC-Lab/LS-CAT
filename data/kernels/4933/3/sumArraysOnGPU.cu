#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx<N)
{
C[idx] = A[idx] + B[idx];
}
}