#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll8Row(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x * 8 + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

unsigned int ti = iy * nx + ix; // access in rows
unsigned int to = ix * ny + iy; // access in columns

if (ix + 7 * blockDim.x < nx && iy < ny)
{
out[to]                   = in[ti];
out[to + ny * blockDim.x]   = in[ti + blockDim.x];
out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
out[to + ny * 4 * blockDim.x] = in[ti + 4 * blockDim.x];
out[to + ny * 5 * blockDim.x] = in[ti + 5 * blockDim.x];
out[to + ny * 6 * blockDim.x] = in[ti + 6 * blockDim.x];
out[to + ny * 7 * blockDim.x] = in[ti + 7 * blockDim.x];
}
}