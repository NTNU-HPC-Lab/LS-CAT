#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;

if (ix<nx && iy<ny)
{
MatC[idx] = sin(MatA[idx]) + sin(MatB[idx]);
}
}