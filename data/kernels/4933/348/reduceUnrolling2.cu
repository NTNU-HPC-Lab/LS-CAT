#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
// set the thread id.
unsigned int tid = threadIdx.x;
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 2;

// convert global data pointer to the local pointer of this block.
int *idata = g_idata + blockIdx.x * blockDim.x * 2;
// unrolling 2 data blocks.
if (idx + blockDim.x < n)
{
g_idata[idx] += g_idata[idx + blockDim.x];
}
__syncthreads();

// boundary check.
if (idx >= n) return;

// in-place reduction in global memory
for (int stride = blockDim.x/2; stride>0; stride>>=1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock.
__syncthreads();
}

// write result for this block to global mem.
if (tid == 0)
{
g_odata[blockIdx.x] = idata[0];
}
}