#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling4 (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 4;

// unrolling 4
if (idx + 3 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4;

// g_idata[idx] = g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2*blockDim.x] + g_idata[idx + 3*blockDim.x];
}

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}