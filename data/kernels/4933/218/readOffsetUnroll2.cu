#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
unsigned int k = i + offset;

if (k < n) C[i] = A[k] + B[k];
if (k + blockDim.x < n) {
C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
}
}