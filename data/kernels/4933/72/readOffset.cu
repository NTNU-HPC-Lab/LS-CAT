#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = i + offset;

if (k < n) C[i] = A[k] + B[k];
}