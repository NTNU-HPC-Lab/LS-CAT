#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling8New (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8
if (idx + 7 * blockDim.x < n)
{
int *ptr = g_idata + idx;
int tmp = 0;

// Increment tmp 8 times with values strided by blockDim.x
for (int i = 0; i < 8; i++) {
tmp += *ptr; ptr += blockDim.x;
}

g_idata[idx] = tmp;
}

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}