#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Row(int *in, int *out, const int nx, const int ny)
{
// set thread id.
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x * 4;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

unsigned int ti = iy * nx + ix; // access in rows.
unsigned int to = ix * ny + iy; // access in cols.

if (ix + 3 * blockDim.x < nx && iy < ny)
{
out[to]                       = in[ti];
out[to + ny * blockDim.x]     = in[ti + blockDim.x];
out[to + ny * blockDim.x * 2] = in[ti + blockDim.x * 2];
out[to + ny * blockDim.x * 3] = in[ti + blockDim.x * 3];
}
}