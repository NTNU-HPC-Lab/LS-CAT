#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
C[i] = A[i] + B[i];
}