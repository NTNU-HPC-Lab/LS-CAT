#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(int *in, int *out, const int nx, const int ny)
{
// set thread id.
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < nx && iy < ny)
{
out[iy * nx + ix] = in[ix * ny + iy];
}
}