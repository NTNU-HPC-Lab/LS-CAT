#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyCol(int *in, int *out, const int nx, const int ny)
{
// set thread id.
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < nx && iy < ny)
{
out[ix * ny + iy] = in[ix * ny + iy];
}
}