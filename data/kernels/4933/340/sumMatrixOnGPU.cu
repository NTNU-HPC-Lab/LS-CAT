#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;

//printf("nx: %d, ny: %d, ix: %d, iy: %d, idx: %d\n", nx, ny, ix, iy, idx);

if (ix<nx && iy<ny)
{
MatC[idx] = MatA[idx] + MatB[idx];
//printf("GPU Add: %f + %f = %f.\n", MatA[idx], MatB[idx], MatC[idx]);
}
}