#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOffset(int *dev_array, int length)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < length)
{
dev_array[tid] += OFFSET;
}
}