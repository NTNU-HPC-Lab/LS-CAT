#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

//=== Vector arithmetic ======================================================

extern "C"


extern "C"


extern "C"


extern "C"

extern "C"




//=== Vector-and-scalar arithmetic ===========================================

extern "C"


extern "C"


extern "C"


extern "C"




extern "C"


extern "C"


extern "C"


extern "C"











//=== Vector comparison ======================================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"



extern "C"




//=== Vector-and-scalar comparison ===========================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"


extern "C"











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"


// Calculate the arc sine of the input argument.
extern "C"


// Calculate the arc hyperbolic sine of the input argument.
extern "C"


// Calculate the arc tangent of the input argument.
extern "C"


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"


// Calculate the cube root of the input argument.
extern "C"


// Calculate ceiling of the input argument.
extern "C"


// Calculate the cosine of the input argument.
extern "C"


// Calculate the hyperbolic cosine of the input argument.
extern "C"


// Calculate the cosine of the input argument × p .
extern "C"


// Calculate the complementary error function of the input argument.
extern "C"


// Calculate the inverse complementary error function of the input argument.
extern "C"


// Calculate the scaled complementary error function of the input argument.
extern "C"


// Calculate the error function of the input argument.
extern "C"


// Calculate the inverse error function of the input argument.
extern "C"


// Calculate the base 10 exponential of the input argument.
extern "C"


// Calculate the base 2 exponential of the input argument.
extern "C"


// Calculate the base e exponential of the input argument.
extern "C"


// Calculate the base e exponential of the input argument, minus 1.
extern "C"


// Calculate the absolute value of its argument.
extern "C"


// Calculate the largest integer less than or equal to x.
extern "C"


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"


// Calculate the base 10 logarithm of the input argument.
extern "C"


// Calculate the value of l o g e ( 1 + x ) .
extern "C"


// Calculate the base 2 logarithm of the input argument.
extern "C"


// Calculate the floating point representation of the exponent of the input argument.
extern "C"


// Calculate the natural logarithm of the input argument.
extern "C"


// Calculate the standard normal cumulative distribution function.
extern "C"


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"


// Calculate reciprocal cube root function.
extern "C"


// Round input to nearest integer value in floating-point.
extern "C"


// Round to nearest integer value in floating-point.
extern "C"


// Calculate the reciprocal of the square root of the input argument.
extern "C"


// Calculate the sine of the input argument.
extern "C"


// Calculate the hyperbolic sine of the input argument.
extern "C"


// Calculate the sine of the input argument × p .
extern "C"


// Calculate the square root of the input argument.
extern "C"


// Calculate the tangent of the input argument.
extern "C"


// Calculate the hyperbolic tangent of the input argument.
extern "C"


// Calculate the gamma function of the input argument.
extern "C"


// Truncate input argument to the integral part.
extern "C"


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"

// Compute the positive difference between x and y.
extern "C"

// Divide two floating point values.
extern "C"

// Determine the maximum numeric value of the arguments.
extern "C"

// Determine the minimum numeric value of the arguments.
extern "C"

// Calculate the floating-point remainder of x / y.
extern "C"

// Calculate the square root of the sum of squares of two arguments.
extern "C"

// Return next representable single-precision floating-point value afer argument.
extern "C"

// Calculate the value of first argument to the power of second argument.
extern "C"

// Compute single-precision floating-point remainder.
extern "C"




__global__ void vec_cbrtf (size_t n, float *result, float  *x)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
if (id < n)
{
result[id] = cbrtf(x[id]);
}
}