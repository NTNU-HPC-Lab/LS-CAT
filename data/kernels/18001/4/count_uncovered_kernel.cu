#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_uncovered_kernel( size_t batch_size, size_t num_nodes, uint32_t *d_rrr_index, uint32_t * d_rrr_sets, char * d_mask, uint32_t *d_counters) {
int pos = threadIdx.x + blockDim.x * blockIdx.x;

if (pos < batch_size) {
uint32_t set = d_rrr_index[pos];
if (d_mask[set] != 1) {
uint32_t v = d_rrr_sets[pos];
atomicAdd(d_counters + v, 1);
}
}
}