#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_mask_kernel( size_t batch_size, uint32_t *d_rrr_index, uint32_t * d_rrr_sets, char *d_mask, uint32_t last_seed) {
size_t pos = threadIdx.x + blockDim.x * blockIdx.x;

if (pos < batch_size && d_rrr_sets[pos] == last_seed) {
uint32_t set = d_rrr_index[pos];
d_mask[set] = 1;
}
}