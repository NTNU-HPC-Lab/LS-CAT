#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_vectors(uint32_t * src, uint32_t * dst, size_t N) {
size_t pos = threadIdx.x + blockDim.x * blockIdx.x;
if (pos < N) {
if (src[pos]) {
atomicAdd(dst + pos, src[pos]);
}
}
}