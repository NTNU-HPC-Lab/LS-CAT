#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
const float beta = *rho * *yDotZ;
*alphaMinusBeta_out = *alpha - beta;
}