#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGradf(const float *d_x, float *d_grad)
{
const float x0 = d_x[0];
const float x1 = d_x[1];

// df/dx0 = -2 (1-x0) - 400 (x1-x0^2) x0
// df/dx1 = 200 (x1 - x0^2)

d_grad[0] = -2.0f * (1.0f - x0) - 400.0f * x0 * (x1 - x0*x0);
d_grad[1] = 200.0f * (x1 - x0*x0);
}