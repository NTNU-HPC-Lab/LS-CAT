#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hyst_kernel(unsigned char *data, unsigned char *out, int rows, int cols) {
// Establish our high and low thresholds as floats
float lowThresh  = 10;
float highThresh = 70;

// These variables are offset by one to avoid seg. fault errors
// As such, this kernel ignores the outside ring of pixels
const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
const int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
const int pos = row * cols + col;

const unsigned char EDGE = 255;

unsigned char magnitude = data[pos];

if(magnitude >= highThresh)
out[pos] = EDGE;
else if(magnitude <= lowThresh)
out[pos] = 0;
else {
float med = (highThresh + lowThresh) / 2;

if(magnitude >= med)
out[pos] = EDGE;
else
out[pos] = 0;
}
}