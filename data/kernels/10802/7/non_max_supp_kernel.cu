#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void non_max_supp_kernel(unsigned char *data, unsigned char *out, unsigned char *theta, int rows, int cols) {

extern __shared__ int l_mem[];
int* l_data = l_mem;

// These variables are offset by one to avoid seg. fault errors
// As such, this kernel ignores the outside ring of pixels
const int L_SIZE = blockDim.x;
const int g_row = blockIdx.y * blockDim.y + threadIdx.y + 1;
const int g_col = blockIdx.x * blockDim.x + threadIdx.x + 1;
const int l_row = threadIdx.y + 1;
const int l_col = threadIdx.x + 1;

const int pos = g_row * cols + g_col;

// copy to l_data
l_data[l_row * (L_SIZE + 2) + l_col] = data[pos];

// top most row
if(l_row == 1) {
l_data[0 * (L_SIZE + 2) + l_col] = data[pos - cols];
// top left
if(l_col == 1)
l_data[0 * (L_SIZE + 2) + 0] = data[pos - cols - 1];

// top right
else if(l_col == L_SIZE)
l_data[0 * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos - cols + 1];
}
// bottom most row
else if(l_row == L_SIZE) {
l_data[(L_SIZE + 1) * (L_SIZE + 2) + l_col] = data[pos + cols];
// bottom left
if(l_col == 1)
l_data[(L_SIZE + 1) * (L_SIZE + 2) + 0] = data[pos + cols - 1];

// bottom right
else if(l_col == L_SIZE)
l_data[(L_SIZE + 1) * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos + cols + 1];
}

if(l_col == 1)
l_data[l_row * (L_SIZE + 2) + 0] = data[pos - 1];
else if(l_col == L_SIZE)
l_data[l_row * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos + 1];

__syncthreads();

unsigned char my_magnitude = l_data[l_row * (L_SIZE + 2) + l_col];

// The following variables are used to address the matrices more easily
switch(theta[pos]) {
// A gradient angle of 0 degrees = an edge that is North/South
// Check neighbors to the East and West
case 0:
// supress me if my neighbor has larger magnitude
if(my_magnitude <= l_data[l_row * (L_SIZE + 2) + l_col + 1] || // east
my_magnitude <= l_data[l_row * (L_SIZE + 2) + l_col - 1]) // west
{
out[pos] = 0;
}
// otherwise, copy my value to the output buffer
else {
out[pos] = my_magnitude;
}
break;

// A gradient angle of 45 degrees = an edge that is NW/SE
// Check neighbors to the NE and SW
case 45:
// supress me if my neighbor has larger magnitude
if(my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col + 1] || // north east
my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col - 1]) // south west
{
out[pos] = 0;
}
// otherwise, copy my value to the output buffer
else {
out[pos] = my_magnitude;
}
break;

// A gradient angle of 90 degrees = an edge that is E/W
// Check neighbors to the North and South
case 90:
// supress me if my neighbor has larger magnitude
if(my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col] || // north
my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col]) // south
{
out[pos] = 0;
}
// otherwise, copy my value to the output buffer
else {
out[pos] = my_magnitude;
}
break;

// A gradient angle of 135 degrees = an edge that is NE/SW
// Check neighbors to the NW and SE
case 135:
// supress me if my neighbor has larger magnitude
if(my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col - 1] || // north west
my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col + 1]) // south east
{
out[pos] = 0;
}
// otherwise, copy my value to the output buffer
else {
out[pos] = my_magnitude;
}
break;

default: out[pos] = my_magnitude; break;
}
}