#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Histogram_kernel(int size, int bins, int cpu_bins, unsigned int *data, unsigned int *histo) {

extern __shared__ unsigned int l_mem[];
unsigned int* l_histo = l_mem;

// Block and thread index
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int bD = blockDim.x;
const int gD = gridDim.x;

// Output partition
int bins_per_wg   = (bins - cpu_bins) / gD;
int my_bins_start = bx * bins_per_wg + cpu_bins;
int my_bins_end   = my_bins_start + bins_per_wg;

// Constants for read access
const int begin = tx;
const int end   = size;
const int step  = bD;

// Sub-histograms initialization
for(int pos = tx; pos < bins_per_wg; pos += bD) {
l_histo[pos] = 0;
}

__syncthreads(); // Intra-block synchronization

// Main loop
for(int i = begin; i < end; i += step) {
// Global memory read
unsigned int d = ((data[i] * bins) >> 12);

if(d >= my_bins_start && d < my_bins_end) {
// Atomic vote in shared memory
atomicAdd(&l_histo[d - my_bins_start], 1);
}
}

__syncthreads(); // Intra-block synchronization

// Merge per-block histograms and write to global memory
for(int pos = tx; pos < bins_per_wg; pos += bD) {
unsigned int sum = 0;
for(int base = 0; base < (bins_per_wg); base += (bins_per_wg))
sum += l_histo[base + pos];
// Atomic addition in global memory
histo[pos + my_bins_start] += sum;
}
}