#include "hip/hip_runtime.h"
#include "includes.h"
/* This code will generate a Sobel image and a Gray Scale image. Uses OpenCV, to compile:
nvcc FinalProject.cu `pkg-config --cflags --libs opencv`

Copyright (C) 2018  Jose Andres Cortez Villao

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.*/



typedef enum color {BLUE, GREEN, RED} Color;	//Constants that contains the values for each color of the image

/*The gray function obtain an average of each pixel and assigned to the correct position in the array using
Channels and step constants*/
/*The sobel function uses a convolution algorithm to obtain the edges of the image */


__global__ void gray(unsigned char *src, unsigned char *dest, int width, int heigth, int step, int channels) {
int ren, col;
float r, g, b;

ren = blockIdx.x; // Variables that parallelize the code
col = threadIdx.x;
r = 0; g = 0; b = 0;

r += (float) src[(ren * step) + (col * channels) + RED];
g += (float) src[(ren * step) + (col * channels) + GREEN];
b += (float) src[(ren * step) + (col * channels) + BLUE];

dest[(ren * step) + (col * channels) + RED] =  (unsigned char) ((r+g+b)/3);
dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) ((r+g+b)/3);
dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) ((r+g+b)/3);
}