#include "hip/hip_runtime.h"
#include "includes.h"
/* This code will generate a Sobel image and a Gray Scale image. Uses OpenCV, to compile:
nvcc FinalProject.cu `pkg-config --cflags --libs opencv`

Copyright (C) 2018  Jose Andres Cortez Villao

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.*/



typedef enum color {BLUE, GREEN, RED} Color;	//Constants that contains the values for each color of the image

/*The gray function obtain an average of each pixel and assigned to the correct position in the array using
Channels and step constants*/
/*The sobel function uses a convolution algorithm to obtain the edges of the image */


__global__ void sobel(unsigned char *src, unsigned char *dest, int width, int heigth, int step, int channels){
int i, j;
int ren, col, tmp_ren, tmp_col;
int gx[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}}; // gx is defined in the Sobel algorithm
int gy[3][3]={{1,2,1},{0,0,0},{-1,-2,-1}}; // gy is defined in the Sobel algorithm
char temp[3][3];

ren = blockIdx.x;
col = threadIdx.x;

tmp_ren = 0;
tmp_col = 0;

//Multiplication of the 3x3 matrix for each color
for (i = -1; i < 2; i++) {
for (j = -1; j < 2; j++) {
temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + RED + i + 1];
tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];
}
}
dest[(ren * step) + (col * channels) + RED] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);;

tmp_ren = 0;
tmp_col = 0;
for (i = -1; i < 2; i++) {
for (j = -1; j < 2; j++) {
temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + GREEN + i + 1];
tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];
}
}
dest[(ren * step) + (col * channels) + GREEN] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);;


tmp_ren = 0;
tmp_col = 0;
for (i = -1; i < 2; i++) {
for (j = -1; j < 2; j++) {
temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + BLUE + i + 1];
tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];
}
}
dest[(ren * step) + (col * channels) + BLUE] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);
}