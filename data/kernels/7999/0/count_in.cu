#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;


#define ULL unsigned long long

const long MAXDIM = 10;
const double RMIN = 2.0;
const double RMAX = 7.0;

#define MAX_THREADS 1024
#define MAX_BLOCKS 65535


//Global kernel code that runs on the device

__global__ void count_in(ULL *dev_count, long dev_ntotal,long dev_ndim, long dev_halfb, double dev_rsquare, long dev_base){

//Calculate the position of this kernel in the data
ULL blockID = (blockIdx.y * gridDim.x) + blockIdx.x;
ULL pos = (blockID * blockDim.x) + threadIdx.x;

//If this threads position in the data is further than we need to calculate
//Then we return
if(pos >= dev_ntotal) return;

double rtestsq = 0;
long idx = 0;
long index[MAXDIM+1];
for (long i = 0; i < dev_ndim; ++i) index[i] = 0;

//Convert the decimal number into another base system
while (pos != 0) {
long rem = pos % dev_base;
pos = pos / dev_base;
index[idx] = rem;
++idx;
}


for(long k = 0; k < dev_ndim; ++k){
double xk = index[k] - dev_halfb;
rtestsq += xk * xk;
}

//If the value is inside the sphere
//Atomically add 1 to the count
if(rtestsq < dev_rsquare){
atomicAdd(dev_count, 1);
}
}