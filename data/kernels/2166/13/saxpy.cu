#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
// Determine our unique global thread ID, so we know which element to process
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int i = tid; i < N; i += stride)
c[i] = 2 * a[i] + b[i];
}