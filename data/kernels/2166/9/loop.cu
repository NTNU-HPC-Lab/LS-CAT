#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void loop()
{
/*
* This idiomatic expression gives each thread
* a unique index within the entire grid.
*/

int i = blockIdx.x * blockDim.x + threadIdx.x;
printf("%d\n", i);
}