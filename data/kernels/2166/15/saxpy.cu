#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
int tid = blockIdx.x * blockDim.x * threadIdx.x;

if (tid < N)
c[tid] = 2 * a[tid] + b[tid];
}