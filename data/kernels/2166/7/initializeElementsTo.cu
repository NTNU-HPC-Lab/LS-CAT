#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
a[i] = initialValue;
}