#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deviceKernel(int *a, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = idx; i < N; i += stride)
{
a[i] = 1;
}
}