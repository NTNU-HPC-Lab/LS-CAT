#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void loop()
{
/*
* This kernel does the work of only 1 iteration
* of the original for loop. Indication of which
* "iteration" is being executed by this kernel is
* still available via `threadIdx.x`.
*/

printf("This is iteration number %d\n", threadIdx.x);
}