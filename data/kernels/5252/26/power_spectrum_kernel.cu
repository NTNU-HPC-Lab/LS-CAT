#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo) {
int thread_id = threadIdx.x;
int block_id = blockIdx.x;
float *Ar = A_in + block_id * ldi;
float *Aw = A_out + block_id * ldo;

int half_length = row_length / 2;
for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
// ignore special case
if (idx == 0) continue;

float2 val = reinterpret_cast<float2 *>(Ar)[idx];
float ret = val.x * val.x + val.y * val.y;
Aw[idx] = ret;
}

// handle special case
if (threadIdx.x == 0) {
float real = Ar[0];
// cufft puts this at the end, this is different than kaldi does with its
// own
// internal implementation
float im = Ar[row_length];

Aw[0] = real * real;
Aw[half_length] = im * im;
}
}