#include "hip/hip_runtime.h"
#include "includes.h"



#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Lazy CUDA Error handling
__global__ void atomicInc_kernel( unsigned int numIterations, unsigned int numInputs, float * d_probabilities, unsigned int * d_quantity, unsigned int * d_count ){
unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

if (tid < numInputs){
if(tid == 0){
printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
}
for (int iteration = 0; iteration < numIterations; iteration++){
// If a value is less than the probabiltiy, apply the min.

unsigned int old = atomicInc(d_quantity + tid, MAX);

// If old is MAX, could not increment.
if(tid == 0){
printf("tid %u: iter %d, old %u\n", tid, iteration, old );
}
if(old < MAX){
d_count[tid]++;
}
}
}
}