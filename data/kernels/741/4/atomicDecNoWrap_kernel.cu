#include "hip/hip_runtime.h"
#include "includes.h"



#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Lazy CUDA Error handling
__device__ unsigned int atomicDecNoWrap(unsigned int * address, unsigned int val){
unsigned int old = *address;
unsigned int assumed;
do {
assumed = old;
old = atomicCAS(address, assumed, (((assumed == 0) | (assumed > val)) ? assumed : (assumed-1)));
} while (assumed != old);
return old;
}
__global__ void atomicDecNoWrap_kernel( unsigned int numIterations, unsigned int numInputs, float * d_probabilities, unsigned int * d_quantity, unsigned int * d_count ){
unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

if (tid < numInputs){
if(tid == 0){
printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
}
for (int iteration = 0; iteration < numIterations; iteration++){

unsigned int old = atomicDecNoWrap(d_quantity + tid, MAX);

if(tid == 0){
printf("tid %u: iter %d, old %u\n", tid, iteration, old );
}

// If old is not the maximum value, we have claimed a resource?
if(old > 0){
d_count[tid]++;
}
}
}
}