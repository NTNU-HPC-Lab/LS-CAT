#include "hip/hip_runtime.h"
#include "includes.h"



#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Lazy CUDA Error handling
__global__ void setQuantities( unsigned int numInputs, unsigned int value, unsigned int * d_quantity ){
unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
if (tid < numInputs){
d_quantity[tid] = value;
}

}