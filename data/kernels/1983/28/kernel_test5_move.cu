#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned int*) end_ptr) {
return;
}

unsigned int half_count = BLOCKSIZE/sizeof(unsigned int)/2;
unsigned int* ptr_mid = ptr + half_count;

for (i = 0;i < half_count; i++){
ptr_mid[i] = ptr[i];
}

for (i=0;i < half_count - 8; i++){
ptr[i + 8] = ptr_mid[i];
}

for (i=0;i < 8; i++){
ptr[i] = ptr_mid[half_count - 8 + i];
}

return;
}