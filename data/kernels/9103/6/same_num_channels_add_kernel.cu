#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void same_num_channels_add_kernel(const float *data_l, const float *data_r, float *result, int total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

if (idx / 2 < total) {
result[idx] = data_l[idx] + data_r[idx];
result[idx + 1] = data_l[idx + 1] + data_r[idx + 1];
}
}