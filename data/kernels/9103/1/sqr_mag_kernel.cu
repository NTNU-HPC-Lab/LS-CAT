#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void sqr_mag_kernel(const float *data, float *result, int total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

if (idx / 2 < total) {
result[idx] = data[idx] * data[idx] + data[idx + 1] * data[idx + 1];
result[idx + 1] = 0;
}
}