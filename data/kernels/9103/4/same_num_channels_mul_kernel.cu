#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void same_num_channels_mul_kernel(const float *data_l, const float *data_r, float *result, int total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

if (idx / 2 < total) {
result[idx] = data_l[idx] * data_r[idx] - data_l[idx + 1] * data_r[idx + 1];
result[idx + 1] = data_l[idx] * data_r[idx + 1] + data_l[idx + 1] * data_r[idx];
}
}