#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
/*
const int sizePoint = 5;
const int sizeIndividum = 5;
const int mathValueMutation = 5;
const float dispersionMutation = 5.0f;
const int powCount = 3;
const float randMaxCount = 20.0f;
*/

const int sizePoint = 500;
const int sizeIndividum = 1000;
const int mathValueMutation = 5;
const float dispersionMutation = 5.0f;
const int powCount = 3;
const float randMaxCount = 20.0f;
const int maxPokoleney = 30;



__global__ void errorsKernel(float *points, float *individs, float *errors, int powCount, int sizePoint)
{

int id = threadIdx.x;
float ans = 0;
int x = 1;
for (int i = 0; i < sizePoint; i++)
{
for (int j = 0; j < powCount; j++)
{
for (int k = 0; k < j; k++)
{
x *= i;
}
x *= individs[id*powCount + j];
ans += x;
x = 1;
}

ans = points[i] - ans;
errors[id] += sqrt(ans * ans);
ans = 0;
}
}