#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mkRender(float *fb, int max_x, int max_y) {
//MK: Pixel 위치 계산을 위해 ThreadId, BlockId를 사용함
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

//MK: 계산된 Pixel 위치가 FB사이즈 보다 크면 연산을 수행하지 않음
if((i >= max_x) || (j >= max_y)){
return;
}

//MK: FB Pixel 값 계산
int pixel_index = j*max_x*3 + i*3;
fb[pixel_index + 0] = float(i) / max_x;
fb[pixel_index + 1] = float(j) / max_y;
fb[pixel_index + 2] = 0.2f;
}