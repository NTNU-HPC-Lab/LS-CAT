#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
const int threads = BLOCK;
__shared__ float local[threads];

int id = threadIdx.x;
local[id] = 0;

int filter = blockIdx.x;

int i, j;
for(j = 0; j < batch; ++j){
for(i = 0; i < spatial; i += threads){
int index = j*spatial*filters + filter*spatial + i + id;
local[id] += (i+id < spatial) ? delta[index] : 0;
}
}
__syncthreads();

if(id == 0){
mean_delta[filter] = 0;
for(i = 0; i < threads; ++i){
mean_delta[filter] += local[i];
}
mean_delta[filter] *= (-1.F/sqrtf(variance[filter] + .000001f));
}
}