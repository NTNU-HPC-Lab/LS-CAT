#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) X[i*INCX] = ALPHA;
}