#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;

x[index] = x[index] - (rate * sqrtf(1.F-powf(B2, t)) / (1.F-powf(B1, t)) * m[index] / (sqrtf(v[index]) + eps));
//if(index == 0) printf("%f %f %f %f\n", m[index], v[index], (rate * sqrtf(1.F-powf(B2, t)) / (1.F-powf(B1, t)) * m[index] / (sqrt(v[index]) + eps)));
}