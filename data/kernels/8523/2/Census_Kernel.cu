#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Census_Kernel(unsigned char * MemSrc, unsigned int * MemDst, int eps, int Width, int Height)
{
//===============================================================================================
//
//===============================================================================================
int globalX = blockIdx.x * blockDim.x + threadIdx.x;
int globalY = blockIdx.y * blockDim.y + threadIdx.y;

int GlobalOffset = (globalY * Width + globalX);
float Value;
float ValueCenter;
unsigned int Census=0;
float Diff = 0;

//int threadX = threadIdx.x+3;
//int threadY = threadIdx.y+3;
//int blockDimX = blockDim.x+2*3;
//int blockDimY = blockDim.y+2*3;

//int OffsetLocal = (threadY * blockDimX + threadX);

extern __shared__ unsigned char DataCache[];
//FillCacheRadius(DataCache, MemSrc, 3, Width, Height);
//------------------------------------------------------------------
if (globalX>1 && globalX<(Width-2) && globalY>1 && globalY<(Height-2))
{
ValueCenter=MemSrc[GlobalOffset];
//ValueCenter=DataCache[OffsetLocal];

#pragma unroll
for(int dy=-1;dy<=1;dy++)
{
#pragma unroll
for(int dx=-1;dx<=1;dx++)
{
if (!(dx==0 && dy==0))
{
Value=MemSrc[(globalY+dy) * Width + (globalX+dx)];
//Value=DataCache[(threadY+dy) * blockDimX + (threadX+dx)];
//---------------------------------------------------------------------
// Ternary
//---------------------------------------------------------------------
Diff = ValueCenter - Value;

Census = Census << 2;

if (abs(Diff)<=eps)
{
Census=Census+1;
}
else if (Diff> eps)
{
Census=Census+2;
}
}
}
}
#pragma unroll
for(int dy=-2;dy<=2;dy++)
{
#pragma unroll
for(int dx=-2;dx<=2;dx++)
{
if (!(dx==0 && dy==0) && !(abs(dx)==1 || abs(dy)==1))
{
Value=MemSrc[(globalY+dy) * Width + (globalX+dx)];
//Value=DataCache[(threadY+dy) * blockDimX + (threadX+dx)];
//---------------------------------------------------------------------
// Ternary
//---------------------------------------------------------------------
Diff = ValueCenter - Value;
Census = Census << 2;

if (abs(Diff)<=eps)
{
Census=Census+1;
}
else if (Diff> eps)
{
Census=Census+2;
}
}
}
}
MemDst[GlobalOffset] = (Census);
}
else
{
if (globalX>=0 && globalX<(Width) && globalY>=0 && globalY<(Height))
MemDst[GlobalOffset] = 0;
}
}