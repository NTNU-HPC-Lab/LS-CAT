#include "hip/hip_runtime.h"
#include "includes.h"
//==========================================================================
// Kernels
//==========================================================================
//==========================================================================
//==========================================================================
// End Kernels
//==========================================================================
//--------------------------------------------------------------------------
//==========================================================================
// Class Methods
//==========================================================================
__global__ void RgbToGray_Kernel(unsigned char * RGB_Image, unsigned char * Gray_Image, int Width, int Height)
{   //------------------------------------------------------------------
int globalX = blockIdx.x * blockDim.x + threadIdx.x;
int globalY = blockIdx.y * blockDim.y + threadIdx.y;
int OffsetGray = (globalY * Width + globalX);
int OffsetColor = (globalY * Width + globalX)*3;
//------------------------------------------------------------------

if(globalX<Width && globalY<Height)
{
Gray_Image[OffsetGray] = (unsigned char)(0.114f*RGB_Image[OffsetColor]+0.587f*RGB_Image[OffsetColor+1]+0.299f*RGB_Image[OffsetColor+2]);
}
}