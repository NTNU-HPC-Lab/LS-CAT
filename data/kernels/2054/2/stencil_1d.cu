#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out){

__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
int gindex = threadIdx.x + blockIdx.x * blockDim.x;
int lindex = threadIdx.x + RADIUS;

// Debugging----------------------
//int *debug_sample = (int *)malloc(3*sizeof(int));

// Read input elements into shared memory
temp[lindex] = in[gindex + RADIUS]; // center

if (threadIdx.x < RADIUS) {
temp[threadIdx.x] = in[gindex]; // left
temp[lindex + BLOCK_SIZE] = in[gindex + RADIUS + BLOCK_SIZE]; // right
}

__syncthreads();

// Apply the stencil
int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++){
result += temp[lindex + offset];
//debug_sample[lindex + offset] = temp[lindex + offset];
}

//Debugging ---------------------
/*printf("Block %d, Thread %d"
" [%d, %d, %d]\n",blockIdx.x,threadIdx.x,
debug_sample[0],debug_sample[1],debug_sample[2]); */

// Store the result
out[gindex] = result;
}