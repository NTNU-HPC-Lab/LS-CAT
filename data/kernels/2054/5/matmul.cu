#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int inner_product(int p, int q, int *a, int c_a, int *b, int r_b, int c_b){
int i = p;
int j = q;
int tmp_ra = i * c_a;
int result = 0;

for(int x = 0; x < r_b; x++){
result += a[tmp_ra] * b[j];
tmp_ra += 1;
j += c_b;
}

return result;
}
__global__ void matmul(int *a, int c_a, int *b, int r_b, int c_b, int *c, int c_c, int N_BLOCKS, int N_THREADS){
int b_indx = blockIdx.x;
int t_indx = threadIdx.x;
int gindex = b_indx * N_THREADS + t_indx;
int i = gindex / c_c;
int j = gindex - i * c_c;
//int gindex = i * c_c  + j;
c[gindex] = inner_product(i, j, a, c_a, b, r_b, c_b);
__syncthreads();
}