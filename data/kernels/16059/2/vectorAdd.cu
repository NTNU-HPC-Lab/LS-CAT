#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const uint16_t* A, const uint16_t* B, uint16_t* C, uint32_t numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
C[i] = A[i] + B[i];
}
}