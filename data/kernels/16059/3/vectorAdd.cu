#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(uint16_t* A, const uint16_t* B, int32_t numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
A[i] += B[i];
}
}