#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void polynomial_expansion (float* poly,int degree,int n,float* array)
{
int INX=blockIdx.x*blockDim.x+threadIdx.x;
if(INX<n)
{
float val=0.0;
float exp=1.0;
for(int x=0;x<=degree;++x)
{
val+=exp*poly[x];
exp*=array[INX];
}
array[INX]=val;
}
}