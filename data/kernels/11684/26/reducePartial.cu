#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

/*
* Perfom a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void reducePartial(int size, void *data, void *result) {
float *fdata = (float*) data;
float *sum = (float*) result;

extern __shared__ float sdata[];

// perform first level of reduction,
// reading from global memory, writing to shared memory unsigned int tid = threadIdx.x;
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = (i < size ? fdata[i] : 0) + (i+blockDim.x < size ? fdata[i+blockDim.x] : 0);
__syncthreads();

// do reduction in shared mem
for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) sum[blockIdx.x] = sdata[0];
}