#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

/*
* Perfom a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void discount(int n, float *a, float *b, float p, float *result)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
result[i] = a[i] / (1.0f + b[i] * p);
}
}