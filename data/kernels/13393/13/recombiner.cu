#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void recombiner( double * rands , unsigned int * parents , unsigned int parent_rows , unsigned int parent_cols , unsigned int * off , unsigned int cols , unsigned int seq_offset ) {
double id_offset = rands[ seq_offset + blockIdx.y ];
__syncthreads();

unsigned int col_offset = (blockIdx.x + threadIdx.y) * blockDim.x + threadIdx.x;

// using integer cast to truncate of fractional portion
unsigned int p0_offset = id_offset * ((parent_rows - 1) / 2);
p0_offset = (2 * p0_offset * parent_cols) + col_offset;

unsigned int p = 0, q = 0, res = 0;
if( col_offset < parent_cols ) {
// should hold true for entire warps
p = parents[ p0_offset ];
q = parents[ p0_offset + parent_cols ];
}
__syncthreads();

if( col_offset < cols ) {
res = off[ (seq_offset + blockIdx.y) * cols + col_offset ];
}
__syncthreads();

res = (( p & ~res ) | ( q & res ));
__syncthreads();

if( col_offset < cols ) {
off[ (seq_offset + blockIdx.y) * cols + col_offset ] = res;
}
}