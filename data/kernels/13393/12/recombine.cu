#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void recombine( unsigned int * p0 , unsigned int * p1 , unsigned int * off , unsigned int cols ) {
unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

unsigned int boffset = blockIdx.x * blockDim.x + tid;

unsigned int p = ((boffset < cols) ? p0[ boffset ] : 0 );
unsigned int q = ((boffset < cols) ? p1[ boffset ] : 0 );
unsigned int res = ((boffset < cols) ? off[ boffset ] : 0 );
__syncthreads();

res = (( p & ~res ) | ( q & res ));
__syncthreads();

if( boffset < cols ) {
off[ boffset ] = res;
}
}