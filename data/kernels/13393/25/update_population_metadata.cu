#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_population_metadata( unsigned int * pop , unsigned int rows , unsigned int cols , unsigned int * free , unsigned int * lost , unsigned int * fixed ) {
unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

__shared__ unsigned int sPop[ MAX_THREADS ];
__shared__ unsigned int sMeta[ MAX_THREADS ];

bool is_fixed_thread = ((threadIdx.y & 1) == 0);
unsigned int eoffset = tid + ((is_fixed_thread) ? blockDim.x : -blockDim.x);
sMeta[ tid ] = ((is_fixed_thread) ? -1 : 0);
__syncthreads();

unsigned int b_offset = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int offset = threadIdx.y * cols + b_offset;
unsigned int mat_size = rows * cols;

unsigned int i = 0;

// scan population
while( i < rows ) {
sPop[ tid ] = ((offset < mat_size) ? pop[offset] : 0 );
__syncthreads();

unsigned int d = sPop[tid], e = sPop[eoffset];
__syncthreads();

unsigned int res = sMeta[ tid ];
if( offset < mat_size ) {
// should only fail when rows is NOT a multiple of THREAD_ROWS
if( is_fixed_thread ) {
res &= (d & e);
} else {
res |= (d | e);
}
}
__syncthreads();

sMeta[ tid ] = res;
__syncthreads();

i += blockDim.y;
offset += blockDim.y * cols;
}

// reduce the fixed and lost lists
i = 4;
while( i <= 32 ) {
unsigned int masked = (threadIdx.y & (i - 1));
unsigned int t =  ((tid + (i / 2) * blockDim.x) & (MAX_THREADS - 1));

// how will branches execute?
// assuming that threads are grouped into warps according to their threadIdx.x coordinate
// all threads in a warp should execute same logic
//
unsigned int res = sMeta[tid], v = sMeta[t];
__syncthreads();

if( masked == 0 ) {
res &= v;
} else if( masked == 1 ) {
res |= v;
}
__syncthreads();

sMeta[ tid ] = res;
__syncthreads();
i <<= 1;
}

// use a single warp to write shared data back to global memory
if( threadIdx.y == 0 ) {
unsigned int fxd = sMeta[ threadIdx.x ];
unsigned int lst = (~sMeta[ blockDim.x + threadIdx.x ]);

free[ b_offset ] = (fxd | lst);
fixed[ b_offset ] = fxd;
lost[ b_offset ] = lst;
}
__syncthreads();
}