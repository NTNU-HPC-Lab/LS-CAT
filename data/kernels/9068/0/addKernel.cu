#include "hip/hip_runtime.h"
#include "includes.h"
//기본 코드


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


//host에서 호출가능하며 Device에서 실행되는 함수 커널함수 정의

//host에서만 호출가능하며 host에서 실행되는 호스트 함수 정의
__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x; // kernel을 실행할 각 thread에게는 thread ID가 주어지는데, kernel 함수 내에서 built-in variable인 ‘threadIdx’로 액세스
c[i] = a[i] + b[i];
printf("%d\n", i);
}