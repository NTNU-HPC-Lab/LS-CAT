#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
float sum = 0;

//행렬에서 계산하려고 하는 위치의 인덱스 이것은 공식화 된것이므로 외우진 말자.
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;


//블록당 쓰레드가 4x4이고
//블록의 개수가 1x1이면
//printf("%d, %d / %d, %d / %d, %d\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
// 4, 4, 0, 0, x, y 이렇게 앞에 4개의 숫자는 고정된 것을 볼 수 있었다.
//blockDim : 블록 안쪽에 포함된 쓰레드가 어떤 ㅁxㅁ 차원으로 되어있는지.
//blockIdx : 블록의 인덱스
//threadIdx : 쓰레드의 인덱스

for (int i = 0; i < arrayWidth; ++i)
{
float Aelement = a[row * arrayWidth + i];
float Belement = b[i*arrayWidth + col];
sum += Aelement * Belement;
}
c[row * arrayWidth + col] = sum;
}