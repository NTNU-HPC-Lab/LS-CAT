#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
float sum = 0;

//여기서 threadIdx.x와 y는 행렬의 인덱스와 같다. 예시) 2x2행렬일때 00 01 10 11

for (int i = 0; i < arrayWidth; ++i)
{
float Aelement = a[threadIdx.y * arrayWidth + i];
float Belement = b[i*arrayWidth + threadIdx.x];
sum += Aelement * Belement;
}
c[threadIdx.y * arrayWidth + threadIdx.x] = sum;
}