#include "hip/hip_runtime.h"
#include "includes.h"








/*
* Naive sort
* used if the quicksort uses too many levels
*/
__global__ void kernel_quicksort(int* values, int n) {
#define MAX_LEVELS	1000

int pivot, L, R;
int idx =  threadIdx.x + blockIdx.x * blockDim.x;
int start[MAX_LEVELS];
int end[MAX_LEVELS];

start[idx] = idx;
end[idx] = n - 1;
while (idx >= 0) {
L = start[idx];
R = end[idx];
if (L < R) {
pivot = values[L];
while (L < R) {
while (values[R] >= pivot && L < R)
R--;
if(L < R)
values[L++] = values[R];
while (values[L] < pivot && L < R)
L++;
if (L < R)
values[R--] = values[L];
}
values[L] = pivot;

start[idx + 1] = L + 1;
end[idx + 1] = end[idx];
end[idx++] = L;


if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1]) {
int tmp = start[idx];
start[idx] = start[idx - 1];
start[idx - 1] = tmp;

tmp = end[idx];
end[idx] = end[idx - 1];
end[idx - 1] = tmp;
}

}
else
idx--;
}
}