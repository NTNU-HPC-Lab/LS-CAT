#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_distance(int* data, float* distance, int* point, int n, int dim) {
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= n)
return;

float d = 0;

for(int j = 0; j<dim; j++)
d += abs(data[i*dim + j] - point[j]);

distance[i] = d;
}