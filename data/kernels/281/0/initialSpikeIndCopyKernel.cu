#include "hip/hip_runtime.h"
#include "includes.h"

/* Error checking */
#define CUDA_ERROR_CHECK
#define CURAND_ERROR_CHECK
#define CUDA_CALL( err) __cudaCall( err, __FILE__, __LINE__ )
#define CURAND_CALL( err) __curandCall( err, __FILE__, __LINE__)
#define CUDA_CHECK_ERROR()    __cudaCheckError( __FILE__, __LINE__ )

__global__ void initialSpikeIndCopyKernel( unsigned short* pLastSpikeInd, const unsigned int noReal)
{
unsigned int globalIndex = threadIdx.x+blockDim.x*blockIdx.x;
unsigned int spikeNo = globalIndex / noReal;
if (globalIndex<noReal*noSpikes)
{
pLastSpikeInd[globalIndex] = pLastSpikeInd[spikeNo*noReal];
}
}