#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialSpikeIndCopyKernel( unsigned short* pLastSpikeInd, const unsigned int noReal)
{
unsigned int globalIndex = threadIdx.x+blockDim.x*blockIdx.x;
unsigned int spikeNo = globalIndex / noReal;
if (globalIndex<noReal*noSpikes)
{
pLastSpikeInd[globalIndex] = pLastSpikeInd[spikeNo*noReal];
}
}