#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandKernel(double* values, int n_original, int factor, double* expanded){
int tid0 = threadIdx.x + blockIdx.x*blockDim.x ;
int stride = blockDim.x*gridDim.x ;
for ( int tid = tid0 ; tid < n_original*factor ; tid += stride){
int idx = floor(double(tid)/factor) ;
expanded[tid] = values[idx] ;
}
}