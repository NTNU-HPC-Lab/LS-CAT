#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void sumByReduction( volatile double* sdata, double mySum, const unsigned int tid )
{
sdata[tid] = mySum;
__syncthreads();

// do reduction in shared mem
if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads();
if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads();

if (tid < 32)
{
sdata[tid] = mySum = mySum + sdata[tid + 32];
sdata[tid] = mySum = mySum + sdata[tid + 16];
sdata[tid] = mySum = mySum + sdata[tid +  8];
sdata[tid] = mySum = mySum + sdata[tid +  4];
sdata[tid] = mySum = mySum + sdata[tid +  2];
sdata[tid] = mySum = mySum + sdata[tid +  1];
}
__syncthreads() ;
}
__global__ void computePdKernel(double* particle_pd, int particles_per_feature, int n_features, double* feature_pd)
{
__shared__ double shmem[256] ;
for ( int n = blockIdx.x ; n < n_features ;n+= gridDim.x ){
int offset = n*particles_per_feature ;
double val = 0 ;
for ( int i = offset+threadIdx.x ; i < offset + particles_per_feature ; i+= blockDim.x ){
val += particle_pd[i] ;
}
sumByReduction(shmem,val,threadIdx.x);

if ( threadIdx.x == 0)
feature_pd[n] = shmem[0]/particles_per_feature ;
__syncthreads() ;
}
}