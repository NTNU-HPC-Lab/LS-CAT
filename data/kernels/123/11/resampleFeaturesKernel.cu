#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resampleFeaturesKernel(double* u, double* v, double* d, double* vu, double* vv, double* vd, double* weights, double* randvals, int n_features, double* u_sampled, double* v_sampled, double* d_sampled, double* vu_sampled, double* vv_sampled, double* vd_sampled)
{
// each block corresponds to 1 feature. there may be more features
// than the maximum number of blocks, so we use this for loop

int n_particles = blockDim.x ;

for ( int n = blockIdx.x ; n < n_features; n += gridDim.x ){
double interval = 1.0/n_particles ;
double r = randvals[n] + threadIdx.x*interval ;

int offset = blockDim.x*n ;
double c = weights[offset] ;
int idx = offset ;
while ( r > c ){
c += weights[++idx] ;

if (idx == offset + n_particles){
idx-- ;
break ;
}
}

int idx_new = n*blockDim.x + threadIdx.x ;
u_sampled[idx_new] = u[idx] ;
v_sampled[idx_new] = v[idx] ;
d_sampled[idx_new] = d[idx] ;
vu_sampled[idx_new] = vu[idx] ;
vv_sampled[idx_new] = vv[idx] ;
vd_sampled[idx_new] = vd[idx] ;
}
}