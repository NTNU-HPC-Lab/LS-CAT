#include "hip/hip_runtime.h"
#include "includes.h"
// CUDA runtime

// nvcc -o cube cube.cu


__global__ void cube(float * d_out, float * d_in){
// Todo: Fill in this function
int idx = threadIdx.x;
float f = d_in[idx];

d_out[idx] = f * f * f;
}