#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000



__global__ void daxpy(int n, double alpha, double *x, double *y) {
for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx < n;
idx += blockDim.x * gridDim.x) {
y[idx] += alpha * x[idx];
}
}