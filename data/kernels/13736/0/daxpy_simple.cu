#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000



__global__ void daxpy_simple(int n, double alpha, double *x, double *y) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < n) {
y[idx] += alpha * x[idx];
}
}