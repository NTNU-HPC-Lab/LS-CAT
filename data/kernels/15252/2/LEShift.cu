#include "hip/hip_runtime.h"
#include "includes.h"
// System includes

// Other c++ libraries

// CUDA runtime

// Helper functions and utilities to work with CUDA

//#define N 2000
#define PI 3.141592653
#define PREC 20
#define maxNeighbors 6
#define maxNeighbors 6
typedef double4 particle;
typedef double dbl;
typedef double3 dbl3;
typedef double2 dbl2;
//typedef float4 particle;
//typedef float2 dbl2;
//typedef float3 dbl3;
//typedef float dbl;

using namespace std;

enum string_code {
enDim,
enumParticles,
ephi,
epotentialPower,
eisFinished,
enone
};

__global__ void LEShift(particle *parts, dbl LEshear) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
parts[i].y = parts[i].y + parts[i].x*LEshear;
return;
}