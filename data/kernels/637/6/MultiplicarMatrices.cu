#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplicarMatrices(float *m1, float *m2, float *mr, int columna1, int fila1, int columna2, int fila2)
{
int fila_r = blockIdx.y*blockDim.y+threadIdx.y;
int columna_r = blockIdx.x*blockDim.x+threadIdx.x;
float tmp_mult = 0;

if ((fila_r < fila2) && (columna_r < columna1)) {
for (int i = 0; i < fila2 ; i++) {
tmp_mult += m1[i+columna1*fila_r]*m2[i*columna2+columna_r];
}

mr[fila_r*columna2+columna_r]= tmp_mult;
}
}