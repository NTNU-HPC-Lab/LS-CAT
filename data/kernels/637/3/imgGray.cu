#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){

int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if ((width > col) && (height > row)){
d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.299+d_image[(row*width+col)*3+1]*0.587+d_image[(row*width+col)*3]*0.114;
}
}