#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;


__global__ void calcpi(int threads, long n, double *results) {
int rank = threadIdx.x;
results[rank] = 0.0;
double step = 1.0/n;
double x = 0.0;

long lower = rank * n/threads;
long upper = (rank + 1) * n/threads;

for (long i = lower; i < upper; i++) {
x    = (i + 0.5) * step;
results[rank] += 4.0 / (1.0 + x*x);
}
}