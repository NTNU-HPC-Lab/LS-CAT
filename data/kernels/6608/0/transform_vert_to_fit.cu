#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void transform_vert_to_fit(const int* src, int* dst, const int nb_vert)
{
const int p = blockIdx.x * blockDim.x + threadIdx.x;
if(p < nb_vert) dst[p] = src[p] < 0 ? 0 : 1;
}