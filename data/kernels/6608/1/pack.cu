#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void pack(const int* prefix_sum, const int* src, int* dst, const int nb_vert)
{
const int p = blockIdx.x * blockDim.x + threadIdx.x;
if(p < nb_vert){
const int elt = src[p];
if(elt >= 0) dst[ prefix_sum[p] ] = elt;
}
}