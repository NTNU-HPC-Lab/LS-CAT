#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void scatter(unsigned int *in,unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int *d_histScan, unsigned int mask, unsigned int current_bits, unsigned int nBins)
{
extern __shared__ unsigned int min_Idx[];

for(int j = threadIdx.x; j < nBins; j += blockDim.x)
min_Idx[j] = n;
__syncthreads();

int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < n)
{
unsigned int bin = (in[i] >> current_bits) & mask;
atomicMin(&min_Idx[bin], i);
}
__syncthreads();

if(i < n)
{
unsigned int bin = (in[i] >> current_bits) & mask;
out[d_histScan[blockIdx.x + bin*gridDim.x] + i - min_Idx[bin]] = in[i];
out_pos[d_histScan[blockIdx.x + bin*gridDim.x] + i - min_Idx[bin]] = in_pos[i];
}
}