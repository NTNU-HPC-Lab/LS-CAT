#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void swap(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < n)
{
in[i] = in[i] ^ out[i];
out[i] = in[i] ^ out[i];
in[i] = in[i] ^ out[i];

in_pos[i] = in_pos[i] ^ out_pos[i];
out_pos[i] = in_pos[i] ^ out_pos[i];
in_pos[i] = in_pos[i] ^ out_pos[i];
}
}