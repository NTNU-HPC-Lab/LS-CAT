#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void pre_sort(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int nBins, unsigned int mask, unsigned int current_bits, unsigned int *d_hist)
{
extern __shared__ unsigned int pre_sort_blk_data[];
unsigned int* blk_value = pre_sort_blk_data;
unsigned int* blk_pos = pre_sort_blk_data + blockDim.x;
unsigned int* blk_hist = pre_sort_blk_data + 2*blockDim.x;
unsigned int* blk_Scan = pre_sort_blk_data + nBins + 2*blockDim.x;

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n)
{
blk_value[threadIdx.x] = in[i];
blk_pos[threadIdx.x] = in_pos[i];
}
__syncthreads();

//Hist
for(int j = threadIdx.x; j < nBins; j += blockDim.x)
{
blk_hist[j] = 0;
blk_Scan[j] = 0;
}
__syncthreads();

unsigned int bin = (blk_value[threadIdx.x] >> current_bits) & mask;
atomicAdd(&blk_hist[bin], 1);
atomicAdd(&blk_Scan[bin], 1);
__syncthreads();

//Scan
for (int stride = 1; stride < nBins; stride *= 2)
{
for (int k = threadIdx.x; k < nBins; k += blockDim.x)
{
int inVal;
if (k >= stride)
inVal = blk_Scan[k - stride];
__syncthreads();
if (k >= stride)
blk_Scan[k] += inVal;
__syncthreads();
}
}
__syncthreads();

for (int i = threadIdx.x; i < nBins; i += blockDim.x)
blk_Scan[i] -= blk_hist[i];
__syncthreads();

//Scatter
if (threadIdx.x == 0)
{
for (int i = 0; i < blockDim.x; i++)
{
unsigned int bin = (blk_value[i] >> current_bits) & mask;
out[blk_Scan[bin] + blockIdx.x*blockDim.x] = blk_value[i];
out_pos[blk_Scan[bin] + blockIdx.x*blockDim.x] = blk_pos[i];
blk_Scan[bin]++;
}
}
}