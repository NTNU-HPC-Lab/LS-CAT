#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void histogram(unsigned int* in, unsigned int* hist, int n,unsigned int nBins, unsigned int mask, unsigned int current_bits)
{
extern __shared__ unsigned int s_local_hist[];

for(int j = threadIdx.x; j < nBins; j += blockDim.x)
s_local_hist[j] = 0;

__syncthreads();

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n)
{
unsigned int bin = (in[i] >> current_bits) & mask;
atomicAdd(&s_local_hist[bin], 1);
}
__syncthreads();

for (int bin = threadIdx.x; bin < nBins; bin += blockDim.x)
{
hist[bin * gridDim.x + blockIdx.x] = s_local_hist[bin];
}
}