#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void exclusive_scan(unsigned int *in,unsigned int *out, int n)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] -= in[i];
}
}