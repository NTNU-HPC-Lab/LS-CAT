#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void addPrevSum(unsigned int* blkSumsScan, unsigned int* blkScans, unsigned int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
if (i < n)
{
blkScans[i] += blkSumsScan[blockIdx.x];
}
}