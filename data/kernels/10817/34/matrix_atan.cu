#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/



/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

extern "C"
__global__ void matrix_atan(double *A, double *C, unsigned int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size){
C[index] = atan(A[index]);
}
}