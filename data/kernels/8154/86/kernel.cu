#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *a, size_t N)
{
int tid = threadIdx.x;
__shared__ float s[BS];
int blocks = (N+BS-1)/BS;
float sum = 0.0f;
for (int ib=0; ib<blocks; ib++)
{
int off = ib*BS+tid;
s[tid] = a[off];
for (int skip=16; skip>0; skip>>=1)
if (tid+skip < N && tid < skip)
s[tid] += s[tid+skip];
sum += s[0];
}
a[0] = sum;
}