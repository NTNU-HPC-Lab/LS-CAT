#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_ker(float* src, float* dst, int N){
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i >= N){
return;
}
dst[i] = fmaxf(0.0, src[i]);
}