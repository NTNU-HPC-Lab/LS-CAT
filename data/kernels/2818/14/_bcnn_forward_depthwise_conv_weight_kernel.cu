#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_forward_depthwise_conv_weight_kernel( int nthreads, float *src_data, float *weight_data, int channels, int dst_h, int dst_w, int src_h, int src_w, int kernel_sz, int stride, int pad, float *dst_data) {
int i, n, c, h, w, kh, kw, h_in, w_in, offset;
float value;
float *weight = NULL;

for (i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads;
i += blockDim.x * gridDim.x) {
n = i / channels / dst_h / dst_w;
c = (i / dst_h / dst_w) % channels;
h = (i / dst_w) % dst_h;
w = i % dst_w;
weight = weight_data + c * kernel_sz * kernel_sz;
value = 0;
for (kh = 0; kh < kernel_sz; ++kh) {
for (kw = 0; kw < kernel_sz; ++kw) {
h_in = -pad + h * stride + kh;
w_in = -pad + w * stride + kw;
if ((h_in >= 0) && (h_in < src_h) && (w_in >= 0) &&
(w_in < src_w)) {
offset = ((n * channels + c) * src_h + h_in) * src_w + w_in;
value += (*weight) * src_data[offset];
}
++weight;
}
}
dst_data[i] = value;
}
}