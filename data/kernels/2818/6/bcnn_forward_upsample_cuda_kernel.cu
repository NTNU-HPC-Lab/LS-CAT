#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_forward_upsample_cuda_kernel(size_t dst_sz, float *src, int w, int h, int c, int n, int size, float *dst) {
size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i >= dst_sz) {
return;
}
int dst_idx = i;
int dst_w = i % (w * size);
i = i / (w * size);
int dst_h = i % (h * size);
i = i / (h * size);
int dst_c = i % c;
i = i / c;
int b = i % n;
int src_w = dst_w / size;
int src_h = dst_h / size;
int src_c = dst_c;

int src_idx = b * w * h * c + src_c * w * h + src_h * w + src_w;

dst[dst_idx] += src[src_idx];
}