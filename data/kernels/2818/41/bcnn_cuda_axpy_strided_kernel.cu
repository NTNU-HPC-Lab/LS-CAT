#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_cuda_axpy_strided_kernel(int n, int num_batches, float a, float *x, float *y, int dst_stride, int src_stride, int x_c, int x_h, int x_w, int y_c, int y_h, int y_w, int min_c, int min_h, int min_w) {
int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) {
return;
}
int i = id % min_w;
id /= min_w;
int j = id % min_h;
id /= min_h;
int k = id % min_c;
id /= min_c;
int b = id % num_batches;

int dst_int = i * dst_stride + y_w * (j * dst_stride + y_h * (y_c * b + k));
int src_ind = i * src_stride + x_w * (j * src_stride + x_h * (x_c * b + k));
y[dst_int] += a * x[src_ind];
}