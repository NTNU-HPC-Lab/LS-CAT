#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "bcnn_cuda_axpy_strided_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int n = XSIZE*YSIZE;
int num_batches = 2;
float a = 2;
float *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
float *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
int dst_stride = 2;
int src_stride = 2;
int x_c = 1;
int x_h = 1;
int x_w = 1;
int y_c = 1;
int y_h = 1;
int y_w = 1;
int min_c = 1;
int min_h = 1;
int min_w = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
bcnn_cuda_axpy_strided_kernel<<<gridBlock,threadBlock>>>(n,num_batches,a,x,y,dst_stride,src_stride,x_c,x_h,x_w,y_c,y_h,y_w,min_c,min_h,min_w);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
bcnn_cuda_axpy_strided_kernel<<<gridBlock,threadBlock>>>(n,num_batches,a,x,y,dst_stride,src_stride,x_c,x_h,x_w,y_c,y_h,y_w,min_c,min_h,min_w);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
bcnn_cuda_axpy_strided_kernel<<<gridBlock,threadBlock>>>(n,num_batches,a,x,y,dst_stride,src_stride,x_c,x_h,x_w,y_c,y_h,y_w,min_c,min_h,min_w);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}