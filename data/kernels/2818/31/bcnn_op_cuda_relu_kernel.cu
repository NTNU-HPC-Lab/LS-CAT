#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_relu_kernel(int n, float *x, float *y) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
y[i] = x[i] * (x[i] > 0);
}
return;
}