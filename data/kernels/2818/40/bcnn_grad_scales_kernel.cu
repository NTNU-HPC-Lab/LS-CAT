#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_grad_scales_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates) {
__shared__ float part[BCNN_CUDA_THREADS];
int i, b;
int filter = blockIdx.x;
int p = threadIdx.x;
float sum = 0;
for (b = 0; b < batch; ++b) {
for (i = 0; i < size; i += BCNN_CUDA_THREADS) {
int index = p + i + size * (filter + n * b);
sum += (p + i < size) ? delta[index] * x_norm[index] : 0;
}
}
part[p] = sum;
__syncthreads();
if (p == 0) {
for (i = 0; i < BCNN_CUDA_THREADS; ++i)
scale_updates[filter] += part[i];
}
}