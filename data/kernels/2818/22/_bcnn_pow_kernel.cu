#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_pow_kernel(int n, float *x, float a, float *y) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) y[i] = pow(x[i], a);
}