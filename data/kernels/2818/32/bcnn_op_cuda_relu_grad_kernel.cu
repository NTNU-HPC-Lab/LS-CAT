#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_relu_grad_kernel(int n, float *x, float *dx) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
dx[i] *= ((float)(x[i] > 0));
}
return;
}