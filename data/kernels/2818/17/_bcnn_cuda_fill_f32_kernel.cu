#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_cuda_fill_f32_kernel(int N, float ALPHA, float *X, int INCX) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < N) X[i * INCX] = ALPHA;
}