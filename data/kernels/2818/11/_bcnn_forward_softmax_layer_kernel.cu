#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_forward_softmax_layer_kernel(int n, int batch, float *input, float *output) {
float sum = 0.f;
float maxf = -INFINITY;
int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

if (b >= batch) {
return;
}
for (int i = 0; i < n; ++i) {
int val = input[i + b * n];
maxf = (val > maxf) ? val : maxf;
}
for (int i = 0; i < n; ++i) {
sum += exp(input[i + b * n] - maxf);
}
sum = (sum != 0) ? maxf + log(sum) : maxf - 100.f;
for (int i = 0; i < n; ++i) {
output[i + b * n] = exp(input[i + b * n] - sum);
}
}