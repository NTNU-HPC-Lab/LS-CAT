#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_backward_upsample_cuda_kernel(size_t dst_sz, float *src, int w, int h, int c, int n, int size, float *dst) {
size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i >= dst_sz) {
return;
}
int dst_idx = i;
int dst_w = i % (w * size);
i = i / (w * size);
int dst_h = i % (h * size);
i = i / (h * size);
int dst_c = i % c;
i = i / c;
int b = i % n;
int in_w = dst_w / size;
int in_h = dst_h / size;
int in_c = dst_c;
int src_idx = b * w * h * c + in_c * w * h + in_h * w + in_w;
src[src_idx] += dst[dst_idx];
}