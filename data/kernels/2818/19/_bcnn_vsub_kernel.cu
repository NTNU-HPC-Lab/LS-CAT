#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_vsub_kernel(int n, float *a, float *b, float *y) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) y[i] = a[i] - b[i];
}