#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mean_variance_forward_kernel(float *x, int b, int c, int wxh, float *mean, float *var) {
float scale = 1.0f / (b * wxh);
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x, j,
k, ind;
if (i >= c) return;

mean[i] = 0;
for (j = 0; j < b; ++j) {
for (k = 0; k < wxh; ++k) {
ind = j * c * wxh + i * wxh + k;
mean[i] += x[ind];
var[i] += x[ind] * x[ind];
}
}
mean[i] *= scale;
var[i] = var[i] * scale - mean[i] * mean[i];
}