#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _norm_backward_kernel(float *x, float *mean, float *var, float *mean_diff, float *var_diff, int b, int c, int wxh, float *grad) {
int ind = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
int j = (ind / wxh) % c;

if (ind >= b * c * wxh) return;

grad[ind] = grad[ind] * 1.0f / (sqrtf(var[j] + 0.00001f)) +
var_diff[j] * 2.0f * (x[ind] - mean[j]) / (wxh * b) +
mean_diff[j] / (wxh * b);
}