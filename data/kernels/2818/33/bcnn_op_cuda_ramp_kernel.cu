#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_ramp_kernel(int n, float *x, float *y) {
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
y[i] = x[i] * (x[i] > 0) + 0.1 * x[i];
}
return;
}