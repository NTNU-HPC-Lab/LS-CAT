#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_backward_depthwise_conv_weight_kernel( int nthreads, float *dst_grad, float *src_data, int batch_size, const int channels, int dst_h, int dst_w, const int src_h, const int src_w, int kernel_sz, int stride, int pad, float *weight_diff) {
int i, n, c, h, w, kw, kh, h_out_s, w_out_s, h_out, w_out, offset;
float *p_weight_diff = NULL;

for (i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads;
i += blockDim.x * gridDim.x) {
n = i / channels / src_h / src_w;
c = (i / src_h / src_w) % channels;
h = (i / src_w) % src_h;
w = i % src_w;
p_weight_diff = weight_diff + c * kernel_sz * kernel_sz;
for (kh = 0; kh < kernel_sz; ++kh) {
for (kw = 0; kw < kernel_sz; ++kw) {
h_out_s = h + pad - kh;
w_out_s = w + pad - kw;
if (((h_out_s % stride) == 0) && ((w_out_s % stride) == 0)) {
h_out = h_out_s / stride;
w_out = w_out_s / stride;
if ((h_out >= 0) && (h_out < dst_h) && (w_out >= 0) &&
(w_out < dst_w)) {
offset = ((n * channels + c) * dst_h + h_out) * dst_w +
w_out;
*p_weight_diff += src_data[i] * dst_grad[offset];
}
}
++p_weight_diff;
}
}
}
}