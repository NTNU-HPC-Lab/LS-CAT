#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_cuda_grad_bias_kernel(float *grad_bias, float *grad_data, int num_channels, int spatial_size) {
int offset = blockIdx.x * blockDim.x + threadIdx.x;
int channel = blockIdx.y;
int batch_size = blockIdx.z;

if (offset < spatial_size)
grad_bias[channel] +=
grad_data[(batch_size * num_channels + channel) * spatial_size +
offset];
}