#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_scales_kernel(float *output, float *biases, int n, int size) {
int offset = blockIdx.x * blockDim.x + threadIdx.x;
int filter = blockIdx.y;
int batch = blockIdx.z;

if (offset < size) {
output[(batch * n + filter) * size + offset] *= biases[filter];
}
}