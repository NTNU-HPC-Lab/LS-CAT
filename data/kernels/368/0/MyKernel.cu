#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MyKernel(int *a, int *b, int *c, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if (idx < N) { c[idx] = a[idx] + b[idx]; }
}