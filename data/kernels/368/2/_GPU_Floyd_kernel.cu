#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _GPU_Floyd_kernel(int k, int *G,int *P, int N){//G will be the adjacency matrix, P will be path matrix
int col=blockIdx.x*blockDim.x + threadIdx.x;
if(col>=N)return;
int idx=N*blockIdx.y+col;

__shared__ int best;
if(threadIdx.x==0)
best=G[N*blockIdx.y+k];
__syncthreads();
if(best==INF || best > 10)return;
int tmp_b=G[k*N+col];
if(tmp_b==INF || tmp_b > 10)return;
//	if (cur > 1)
//		return;
int cur = best + tmp_b;
if(cur<G[idx]){
G[idx]=cur;
P[idx]=k;
}
}