#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000

__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
__shared__ unsigned int cache[256];
int i = blockIdx.x * blockDim.x + threadIdx.x;

cache[threadIdx.x] = 0;
__syncthreads();

while(i < N)
{
atomicAdd(&cache[data[i]], 1);
i += blockDim.x * gridDim.x;
}

__syncthreads();
atomicAdd(&histogram[threadIdx.x], cache[threadIdx.x]);
}