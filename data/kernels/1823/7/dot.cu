#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int *a, int *b, int *c)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
while(i < N)
{
c[i] = a[i] * b[i];
i += blockDim.x * gridDim.x;
}
}