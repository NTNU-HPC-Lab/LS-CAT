#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int *a, int *b, int *temp, int *c)
{
int outputIndex = blockIdx.x * blockDim.x + threadIdx.x;
int i = outputIndex;
int result = 0;

/* multiplication step: compute partial sum */
while(i < N)
{
result += a[i] * b[i];
i += blockDim.x * gridDim.x;
}

temp[outputIndex] = result;

/* wait for all threads to be done multiplying */
__syncthreads();

/* reduction step: sum all entries in the block and write to c */
/* this requires that blockDim.x be a power of two! */
i = blockDim.x / 2;
while (i != 0)
{
/* only threads 0 through i are busy */
if (threadIdx.x < i)
{
/* sum our output element with the one half a block away */
temp[outputIndex] += temp[outputIndex + i];
}

/* wait for all threads within the block */
__syncthreads();

i /= 2;
}

/* thread 0 writes the results for this block */
if (threadIdx.x == 0)
{
c[blockIdx.x] = temp[outputIndex];
}
}