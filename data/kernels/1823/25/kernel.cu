#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( uchar4 *ptr, int ticks ) {
// map from threadIdx/BlockIdx to pixel position
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

// now calculate the value at that position
float fx = x - DIM/2;
float fy = y - DIM/2;
float d = sqrtf( fx * fx + fy * fy );
unsigned char grey = (unsigned char)(128.0f + 127.0f *
cos(d/10.0f - ticks/7.0f) /
(d/10.0f + 1.0f));
ptr[offset].x = grey;
ptr[offset].y = grey;
ptr[offset].z = grey;
ptr[offset].w = 255;
}