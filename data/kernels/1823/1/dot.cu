#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int *a, int *b, int *c)
{
/* shared memory cache for partial sum results */
__shared__ int cache[THREADS_PER_BLOCK];

int i = blockIdx.x * blockDim.x + threadIdx.x;
int result = 0;

/* multiplication step: write a partial sum into the cache */
while(i < N)
{
result += a[i] * b[i];
i += blockDim.x * gridDim.x;
}

cache[threadIdx.x] = result;

/* wait for all other threads in the same block */
__syncthreads();

/* reduction step: sum all entries in the cache */
i = blockDim.x / 2;
while (i != 0)
{
/* only threads 0 through i are busy */
if (threadIdx.x < i)
{
cache[threadIdx.x] += cache[threadIdx.x + i];
}

/* wait for all threads within the block */
__syncthreads();

i /= 2;
}

/* thread 0 writes the result for this block */
if (threadIdx.x == 0)
{
c[blockIdx.x] = cache[0];
}
}