#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a, int *b, int *c)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
c[i] = a[i] + b[i];
}