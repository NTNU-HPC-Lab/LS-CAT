#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {

// clear out the accumulation buffer called temp
// since we are launched with 256 threads, it is easy
// to clear that memory with one write per thread
__shared__  unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

// calculate the starting index and the offset to the next
// block that each thread will be processing
int i = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while (i < size) {
atomicAdd( &temp[buffer[i]], 1 );
i += stride;
}
// sync the data from the above writes to shared memory
// then add the shared memory values to the values from
// the other thread blocks using global memory
// atomic adds
// same as before, since we have 256 threads, updating the
// global histogram is just one write per thread!
__syncthreads();
atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}