#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

while(i < N)
{
atomicAdd(&histogram[data[i]], 1);
i += blockDim.x * gridDim.x;
}
}