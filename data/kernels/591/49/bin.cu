#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bin(unsigned short *d_input, float *d_output, int in_nsamp) {

int c = ( ( blockIdx.y * BINDIVINF ) + threadIdx.y );
int out_nsamp = ( in_nsamp ) / 2;
int t_out = ( ( blockIdx.x * BINDIVINT ) + threadIdx.x );
int t_in = 2 * t_out;

size_t shift_one = ( (size_t)(c*out_nsamp) + (size_t)t_out );
size_t shift_two = ( (size_t)(c*in_nsamp)  + (size_t)t_in );

d_output[( shift_one )] = (float) ( ( d_input[( shift_two )] + d_input[(size_t)(shift_two + 1)] )/2.0f );

}