#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

size_t t = blockIdx.x * blockDim.x + threadIdx.x;
size_t c = blockIdx.y * blockDim.y + threadIdx.y;

d_output[(size_t)(c * nsamp) + t] = (float) __ldg(&d_input[(size_t)(t * nchans) + c]);

}