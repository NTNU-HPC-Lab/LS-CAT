#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_stretch_kernel(int samps, float mean, float *d_input) {

int t = blockIdx.x * blockDim.x + threadIdx.x;

if (t >= 0 && t < samps)
d_input[t] = mean;
}