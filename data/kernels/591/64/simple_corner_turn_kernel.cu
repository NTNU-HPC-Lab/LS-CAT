#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size){

size_t primary = blockIdx.x * blockDim.x + threadIdx.x;
size_t secondary = blockIdx.y * blockDim.y + threadIdx.y;

d_output[(size_t)primary*secondary_size + secondary] = (float) __ldg(&d_input[(size_t)secondary*primary_size + primary]);
}