#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bootstrap(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
float myResample = 0.0f;

unsigned int constant = ( 4294967295 / ( bins - blockDim.x ) );
int constant2 = blockIdx.x * bins;
int dmid = bins * ( blockDim.y * blockIdx.y + threadIdx.y );
for (int i = 0; i < bins; i++)
{
int rid = ( g_irand[constant2 + i] / ( constant ) );
myResample += g_idata[rid + dmid + threadIdx.x];
}
dmid = num_boots * ( blockDim.y * blockIdx.y + threadIdx.y );
g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ( (double) myResample / (double) num_els );
}