#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_Filter_peaks_kernel(unsigned int *d_new_peak_list_DM, unsigned int *d_new_peak_list_TS, unsigned int *d_new_peak_list_BW, float *d_new_peak_list_SNR, unsigned int *d_peak_list_DM, unsigned int *d_peak_list_TS, unsigned int *d_peak_list_BW, float *d_peak_list_SNR, unsigned int nElements, unsigned int max_distance, int nLoops, int max_list_pos, int *gmem_pos){
// PPF_DPB = 128 //this is because I set nThreads to 64
// PPF_PEAKS_PER_BLOCK = something small like 10
__shared__ float s_data_snr[PPF_DPB];
__shared__ int s_data_dm[PPF_DPB];
__shared__ int s_data_ts[PPF_DPB];
__shared__ int s_flag[PPF_NTHREADS];
int d, s;
int elements_pos, pos;
float snr, distance, fs, fd;
//	float4 f4temp;


if(threadIdx.x<PPF_PEAKS_PER_BLOCK){
s_flag[threadIdx.x] = 1;
}
else{
s_flag[threadIdx.x] = 0;
}


for(int f=0; f<nLoops; f++){
// Load new data blob
//s_data[threadIdx.x + 2*PPF_DPB] = 0; // SNR
//s_data[threadIdx.x + 64 + 2*PPF_DPB] = 0; // SNR

pos = PPF_DPB*f + threadIdx.x;
if(pos < nElements){
//			f4temp = __ldg(&d_peak_list[pos]);
s_data_dm[threadIdx.x]  = d_peak_list_DM[pos]; //f4temp.x; // DM
s_data_ts[threadIdx.x]  = d_peak_list_TS[pos]; //f4temp.y; // Time
s_data_snr[threadIdx.x] = d_peak_list_SNR[pos]; //f4temp.z; // SNR
}
else {
s_data_dm[threadIdx.x]  = 0; //f4temp.x; // DM
s_data_ts[threadIdx.x]  = 0; //f4temp.y; // Time
s_data_snr[threadIdx.x] = -1000; //f4temp.z; // SNR
}
//		if(blockIdx.x==0 && threadIdx.x==0) printf("point: [%d;%d;%lf]\n",  s_data_dm[threadIdx.x], s_data_ts[threadIdx.x], s_data_snr[threadIdx.x]);


pos = PPF_DPB*f + threadIdx.x + PPF_NTHREADS;
if(pos < nElements){
//			f4temp = __ldg(&d_peak_list[PPF_DPB*f + threadIdx.x + (PPF_DPB>>1)]);
s_data_dm[threadIdx.x + PPF_NTHREADS ] = d_peak_list_DM[pos]; //f4temp.x; // DM
s_data_ts[threadIdx.x + PPF_NTHREADS ] = d_peak_list_TS[pos]; //f4temp.y; // Time
s_data_snr[threadIdx.x + PPF_NTHREADS] = d_peak_list_SNR[pos]; //f4temp.z; // SNR
}
else {
s_data_dm[threadIdx.x + PPF_NTHREADS]  = 0; //f4temp.x; // DM
s_data_ts[threadIdx.x + PPF_NTHREADS]  = 0; //f4temp.y; // Time
s_data_snr[threadIdx.x + PPF_NTHREADS] = -1000; //f4temp.z; // SNR
}

__syncthreads();

elements_pos = blockIdx.x*PPF_PEAKS_PER_BLOCK;
for(int p=0; p<PPF_PEAKS_PER_BLOCK; p++){
//			if (blockIdx.x == 0) printf("%d %d\n", p, s_flag[p]);
if((s_flag[p]) && ((elements_pos + p) < nElements)){
//pos = elements_pos+p;
//if(pos<nElements){
d   = d_peak_list_DM[elements_pos+p]; // DM
s   = d_peak_list_TS[elements_pos+p]; // Time
snr = d_peak_list_SNR[elements_pos+p]; // SNR

// first element
//					if(blockIdx.x==0) printf("s_data: %lf, snr: %lf, pos: %d\n", s_data_snr[threadIdx.x], snr, p);
if( (s_data_snr[threadIdx.x] >= snr)){
fs = ((float)s_data_dm[threadIdx.x] - (float)d);
fd = ((float)s_data_ts[threadIdx.x] - (float)s);
distance = (fd*fd + fs*fs);
//						if(blockIdx.x==0) printf("%d - %d = %d; %d - %d = %d\n",s_data_dm[threadIdx.x], d, fs, s_data_ts[threadIdx.x], s, fd, distance);
if( (distance < (float)max_distance) && (distance!=0) ){
//							if(blockIdx.x==0) printf("distance: %d %lf %lf %lf %d %d;\n", p, distance, fs, fd, s, d);
s_flag[p]=0;
}
}

//second element
if(s_data_snr[threadIdx.x + PPF_NTHREADS] >= snr){
fs = ((float)s_data_dm[threadIdx.x + PPF_NTHREADS] - (float)d);
fd = ((float)s_data_ts[threadIdx.x + PPF_NTHREADS] - (float)s);
distance = (fd*fd + fs*fs);
//						if(blockIdx.x==0) printf("%d - %d = %d; %d - %d = %d\n",s_data_dm[threadIdx.x], d, fs, s_data_ts[threadIdx.x], s, fd, distance);
if( (distance < (float)max_distance) && (distance!=0)){
s_flag[p]=0;
//							if(blockIdx.x==0) printf("xdistance: %d %lf %lf %lf %d %d;\n", p, distance, fs, fd, s, d);
}
}
//}
}
} // for p

}

// Saving peaks that got through
elements_pos = blockIdx.x*PPF_PEAKS_PER_BLOCK;
if(threadIdx.x < PPF_PEAKS_PER_BLOCK){
if( (s_flag[threadIdx.x] == 1) && ((elements_pos + threadIdx.x) < nElements)){
int list_pos=atomicAdd(gmem_pos, 1);
if(list_pos<max_list_pos){
d_new_peak_list_DM[list_pos]  = d_peak_list_DM[elements_pos  + threadIdx.x];
d_new_peak_list_TS[list_pos]  = d_peak_list_TS[elements_pos  + threadIdx.x];
d_new_peak_list_BW[list_pos]  = d_peak_list_BW[elements_pos  + threadIdx.x];
d_new_peak_list_SNR[list_pos] = d_peak_list_SNR[elements_pos + threadIdx.x];
}
}
}
}