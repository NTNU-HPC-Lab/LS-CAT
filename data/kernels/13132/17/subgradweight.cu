#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
// iterators
int xx, yy;

// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;
int k = blockIdx.x % input_n;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y;

// select input/output plane
gradOutput = gradOutput + o*output_w*output_h;
input = input + i*input_w*input_h;

// thread ID
int tid = blockDim.x*threadIdx.y + threadIdx.x;

// create array to hold partial sums
__shared__ float sums[CUDA_MAX_THREADS];
sums[tid] = 0;

// compute partial sums
for(yy = yy_start; yy < yy_end; yy+=yy_step) {
for(xx = xx_start; xx < xx_end; xx+=xx_step) {
float *ptr_input = input + yy*dH*input_w + xx*dW;
float *ptr_gradOutput = gradOutput + yy*output_w + xx;
float z = *ptr_gradOutput;
long kx, ky;
for(ky = 0; ky < kH; ky++) {
for(kx = 0; kx < kW; kx++) {
sums[tid] += z * ptr_input[kx];
}
ptr_input += input_w;
}
}
}
__syncthreads();

// reduce: accumulate all partial sums to produce final gradWeight
if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
for(int i = 0; i < blockDim.x*blockDim.y; i++) gradWeight[k] += scale*sums[i];
}
__syncthreads();

// compute gradBias
sums[tid] = 0;
for (int i=tid; i<output_w*output_h; i+=(blockDim.x*blockDim.y)) {
sums[tid] += gradOutput[i];
}
__syncthreads();

// reduce gradBias
if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
for (int i=0; i<(blockDim.x*blockDim.y); i++)
gradBias[k] += scale*sums[i];
}
}