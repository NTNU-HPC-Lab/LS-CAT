#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imagePaddingKernel(float3 *ptr, float3 *dst, int width, int height, int top, int bottom, int left, int right)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
if(x < left || x >= (width - right) || y < top || y > (height - bottom)) {
return;
}

float3 color = ptr[(y - top) * (width - top - right) + (x - left)];

dst[y * width + x] = color;
}