#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertBGR2RGBfloatKernel(uchar3 *src, float3 *dst, int width, int height)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
if (x >= width || y >= height) {
return;
}

uchar3 color = src[y * width + x];
dst[y * width + x] = make_float3(color.z, color.y, color.x);
}