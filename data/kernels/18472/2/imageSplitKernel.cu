#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageSplitKernel(float3 *ptr, float *dst, int width, int height)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
if (x >= width || y >= height) {
return;
}

float3 color = ptr[y * width + x];

dst[y * width + x] = color.x;
dst[y * width + x + width * height] = color.y;
dst[y * width + x + width * height * 2] = color.z;
}