#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcPReLUKernel(const float *input, float *output, const float *weights, int width, int height, int channels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
if (x >= width || y >= height) {
return;
}

output[y * width + x] = input[y * width + x] > 0 ? input[y * width + x] : input[y * width + x] * weights[y % channels];

}