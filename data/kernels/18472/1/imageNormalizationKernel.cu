#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageNormalizationKernel(float3 *ptr, int width, int height)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
if (x >= width || y >= height) {
return;
}

float3 color = ptr[y * width + x];
color.x = (color.x - 127.5) * 0.0078125;
color.y = (color.y - 127.5) * 0.0078125;
color.z = (color.z - 127.5) * 0.0078125;

ptr[y * width + x] = make_float3(color.x, color.y, color.z);
}