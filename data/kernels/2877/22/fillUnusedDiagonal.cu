#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillUnusedDiagonal(float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
{
int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

if (sampleIdx < numSamples)
{
int subArraySizeLocal = subArraySizes[sampleIdx];
int numelRfull = subArraySize * subArraySize;

if (subArraySize > subArraySizeLocal)
{
float* R = &Rmatrices[sampleIdx*numelRfull];
float diagEntry = R[subArraySize*subArraySize - 1];

for (int diagIdx = subArraySizeLocal + tIdx; diagIdx < subArraySize; diagIdx += blockDim.x*blockDim.y)
{
// subArraySize + 1 (instead of subArraySize) to follow the diagonal
int matrixIdx = diagIdx * (subArraySize + 1);

R[matrixIdx] = diagEntry;
}
}
}
}