#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;

float mhat = m[index] / (1.f - powf(B1, t));
float vhat = v[index] / (1.f - powf(B2, t));

x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}