#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
int h = (in_h + pad - size)/stride + 1;
int w = (in_w + pad - size)/stride + 1;
int c = in_c;
int area = (size-1)/stride;

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= n) return;

int index = id;
int j = id % in_w;
id /= in_w;
int i = id % in_h;
id /= in_h;
int k = id % in_c;
id /= in_c;
int b = id;

int w_offset = -pad/2;
int h_offset = -pad/2;

float d = 0;
int l, m;
for(l = -area; l < area+1; ++l){
for(m = -area; m < area+1; ++m){
int out_w = (j-w_offset)/stride + m;
int out_h = (i-h_offset)/stride + l;
int out_index = out_w + w*(out_h + h*(k + c*b));
int valid = (out_w >= 0 && out_w < w &&
out_h >= 0 && out_h < h);
d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
}
}
prev_delta[index] += d;
}