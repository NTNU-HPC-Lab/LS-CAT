#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void differenceImg(float *d_Octave0,float *d_Octave1,float *d_diffOctave,int pitch,int height){

int x = blockIdx.x*blockDim.x+threadIdx.x;
int y = blockIdx.y*blockDim.y+threadIdx.y;

int index = y * pitch + x;
if(y<height)
d_diffOctave[index] = (d_Octave1[index] - d_Octave0[index]);

}