#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void addpoint(){

}
__device__ unsigned int d_PointCounter[1]; //choose 60 suport 16384 pixel size image (log2(16384) - 2)*5
__device__ float *pd[60];
//choose 72 suport 16384 pixel size image (log2(16384) - 2)*6
__device__ float *pgpyr[72];
__device__ int temsize;
//36 suppose the max Octave is 12
__constant__ int d_oIndex[36];

static const int BLOCK_SIZE_ONE_DIM = 32;
static const int Descript_BLOCK_SIZE = 32;

__global__ void test_gpu(int pitch,int height)
{
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if(y<height)
{
pgpyr[0][y*pitch+x] += 100;
}
if(y*pitch+x<5)
printf("%f\n",pd[0][y*pitch+x]);
}
__global__ void findScaleSpaceExtrema_gpu(float *d_point,int p_pitch,int s, int width ,int pitch ,int height,const int threshold,const int nOctaveLayers,const int maxNum){

int x = blockIdx.x*blockDim.x+threadIdx.x;
int y = blockIdx.y*blockDim.y+threadIdx.y;

//avoid extract the unstable border points
if(y >= height - SIFT_IMG_BORDER || x >= width - SIFT_IMG_BORDER || x<SIFT_IMG_BORDER || y<SIFT_IMG_BORDER)
return;

float *currptr = pd[s]  +y*pitch+x;
float *prevptr = pd[s-1]+y*pitch+x;
float *nextptr = pd[s+1]+y*pitch+x;

int o = s/(nOctaveLayers+2);
float val = *currptr;
int step = pitch;
int c = 0;
if( std::abs(val) > threshold &&
((val > 0 && val >= currptr[c-1] && val >= currptr[c+1] &&
val >= currptr[c-step-1] && val >= currptr[c-step] && val >= currptr[c-step+1] &&
val >= currptr[c+step-1] && val >= currptr[c+step] && val >= currptr[c+step+1] &&
val >= nextptr[c] && val >= nextptr[c-1] && val >= nextptr[c+1] &&
val >= nextptr[c-step-1] && val >= nextptr[c-step] && val >= nextptr[c-step+1] &&
val >= nextptr[c+step-1] && val >= nextptr[c+step] && val >= nextptr[c+step+1] &&
val >= prevptr[c] && val >= prevptr[c-1] && val >= prevptr[c+1] &&
val >= prevptr[c-step-1] && val >= prevptr[c-step] && val >= prevptr[c-step+1] &&
val >= prevptr[c+step-1] && val >= prevptr[c+step] && val >= prevptr[c+step+1]) ||
(val < 0 && val <= currptr[c-1] && val <= currptr[c+1] &&
val <= currptr[c-step-1] && val <= currptr[c-step] && val <= currptr[c-step+1] &&
val <= currptr[c+step-1] && val <= currptr[c+step] && val <= currptr[c+step+1] &&
val <= nextptr[c] && val <= nextptr[c-1] && val <= nextptr[c+1] &&
val <= nextptr[c-step-1] && val <= nextptr[c-step] && val <= nextptr[c-step+1] &&
val <= nextptr[c+step-1] && val <= nextptr[c+step] && val <= nextptr[c+step+1] &&
val <= prevptr[c] && val <= prevptr[c-1] && val <= prevptr[c+1] &&
val <= prevptr[c-step-1] && val <= prevptr[c-step] && val <= prevptr[c-step+1] &&
val <= prevptr[c+step-1] && val <= prevptr[c+step] && val <= prevptr[c+step+1])))
{
/*adjustLocalExtrema*/
const float img_scale = 1.f/(255*SIFT_FIXPT_SCALE);
const float deriv_scale = img_scale*0.5f;
const float second_deriv_scale = img_scale;
const float cross_deriv_scale = img_scale*0.25f;
float Vs=0, Vx=0, Vy=0, contr=0;
float dx,dy,ds,dxx,dyy,dxy;
int j = 0,layer;
//get the x,y,s,Vs,Vx,Vy or return
for( ; j < SIFT_MAX_INTERP_STEPS; j++ )
{
currptr = pd[s]  +y*pitch+x;
prevptr = pd[s-1]+y*pitch+x;
nextptr = pd[s+1]+y*pitch+x;

//the first derivative of x,y and scale
dx = (currptr[1] - currptr[-1])*deriv_scale;
dy = (currptr[pitch] - currptr[-pitch])*deriv_scale;;
ds = (nextptr[0] - prevptr[0])*deriv_scale;
float v2 = currptr[0]*2;

//the second derivative of x,y,scale
dxx = (currptr[1] + currptr[-1] - v2)*second_deriv_scale;
dyy = (currptr[pitch] + currptr[-pitch] - v2)*second_deriv_scale;
float dss = (nextptr[0] + prevptr[0] - v2)*second_deriv_scale;
dxy = (currptr[pitch+1] - currptr[1-pitch] -
currptr[-1+pitch] + currptr[-pitch-1])*cross_deriv_scale;
float dxs = (nextptr[1] - nextptr[-1] -
prevptr[1] + prevptr[-1])*cross_deriv_scale;
float dys = (nextptr[pitch] - nextptr[-pitch] -
prevptr[pitch] + prevptr[-pitch])*cross_deriv_scale;

//Algebraic cousin
float idxx = dyy*dss - dys*dys;
float idxy = dys*dxs - dxy*dss;
float idxs = dxy*dys - dyy*dxs;
//idet is the det,the matrix's determinant countdown
float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
float idyy = dxx*dss - dxs*dxs;
float idys = dxy*dxs - dxx*dys;
float idss = dxx*dyy - dxy*dxy;
////////////////////////
///  A(dxx, dxy, dxs,
///    dxy, dyy, dys,
///    dxs, dys, dss);
///
///  A*(idxx, idxy, idxs,
///     idxy, idyy, idys,
///     idxs, idys, idss);
///
///  B(dx,dy,dz)
/////////////////////////
//dX = (A^-1)*B
float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
float pds = idet*(idxs*dx + idys*dy + idss*ds);

Vx = -pdx;
Vy = -pdy;
Vs = -pds;

//because of the judgment is before the updated value,so
//this iteration final get the x,y,s(intger) and the Vx,Vy,Vz(<0.5).
//The accurate extrema location is x+Vx,y+Vy.

if( std::abs(Vs) < 0.5f && std::abs(Vx) < 0.5f && std::abs(Vy) < 0.5f )
break;

//get nearest intger
x += int(Vx > 0 ? ( Vx + 0.5 ) : (Vx - 0.5));
y += int(Vy > 0 ? ( Vy + 0.5 ) : (Vy - 0.5));
s += int(Vs > 0 ? ( Vs + 0.5 ) : (Vs - 0.5));

layer = s - o*(nOctaveLayers+2);

if( layer < 1 || layer > nOctaveLayers ||
y < SIFT_IMG_BORDER || y >= height - SIFT_IMG_BORDER  ||
x < SIFT_IMG_BORDER || x >= width - SIFT_IMG_BORDER )
return;

}//for
if( j >= SIFT_MAX_INTERP_STEPS )
return;

//After the iterative,get the x,y,s,(Vx,Vy,Vs)(<0.5).

{
//remove the small energy points which essily influenced by image noise
float t = dx*Vx + dy*Vy + ds*Vs;
contr = currptr[0]*img_scale + t * 0.5f;
if( std::abs( contr ) * nOctaveLayers < 0.04 )
return;

// principal curvatures are computed using the trace and det of Hessian
float tr = dxx + dyy;
float det = dxx*dyy-dxy*dxy;

if( det <= 0 || tr*tr*10 >= (10 + 1)*(10 + 1)*det )
return;
}

layer = s - o*(nOctaveLayers+2);

//addpoint;
unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
idx = (idx>=maxNum ? maxNum-1 : idx);
//        d_point[idx*KEYPOINTS_SIZE] = (x + Vx)*(1 << o);
//        d_point[idx*KEYPOINTS_SIZE+1] = (y + Vy)*(1 << o);
//        d_point[idx*KEYPOINTS_SIZE+2] = o + (s<<8) + ((int)(((Vs + 0.5)*255)+0.5) << 16);
//        float size = 1.6*__powf(2.f, (layer + Vs) / nOctaveLayers)*(1 << o)*2;
//        d_point[idx*KEYPOINTS_SIZE+3] = size;
//        d_point[idx*KEYPOINTS_SIZE+4] = std::abs(contr);
//        d_point[idx*KEYPOINTS_SIZE+6] = s;
//        d_point[idx*KEYPOINTS_SIZE+7] = x;
//        d_point[idx*KEYPOINTS_SIZE+8] = y;

d_point[idx] = (x + Vx)*(1 << o);
d_point[idx+p_pitch*1] = (y + Vy)*(1 << o);
float oct_lay1 =o + (layer<<8) + ((int)(((Vs + 0.5)*255)+0.5) << 16);
//        if(idx == 0)
//            printf("o = %d ,Vs = %f ,oct_lay1 = %f ,x = %f , y = %f\n",o,Vs,oct_lay1,d_point[idx],d_point[idx+p_pitch*1]);
//int oct_lay = oct_lay1;
d_point[idx+p_pitch*2] = oct_lay1;
float size = 1.6*__powf(2.f, (layer + Vs) / nOctaveLayers)*(1 << o)*2;
d_point[idx+p_pitch*3] = size;
d_point[idx+p_pitch*4] = std::abs(contr);
//        int _octave,_layer;
//        _octave = oct_lay & 255;
//        layer = (oct_lay >> 8) & 255;
//        _octave = _octave < 128 ? _octave : (-128 | _octave);
//        s = _octave*(nOctaveLayers+2)+layer;
//        x = round(d_point[idx]/(1<<_octave));
//        y = round(d_point[idx+p_pitch*1]/(1<<_octave));
//        d_point[idx+p_pitch*6] = s;
//        d_point[idx+p_pitch*7] = x;
//        d_point[idx+p_pitch*8] = y;

//temsize+=size*0.5f/(1 << o)*SIFT_ORI_RADIUS+0.5;
//        if(x<2000 && y<2000)
//            printf("%d,%d,%d\n",x,y,s);
//printf("%f \n",pd[0][100*2304+100]);

//        float scl_octv = size*0.5f/(1 << o);
//        //'+0.5' for rounding because scl_octv>0
//        int radius = SIFT_ORI_RADIUS * scl_octv+0.5;
//        //the procress of all point range, a square space.
//        int len = (radius*2+1)*(radius*2+1);
//        //int temBuffSize = len*4+2*SIFT_ORI_HIST_BINS+2;
//        atomicMax(&temsize,len);
}
}