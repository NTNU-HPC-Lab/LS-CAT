#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply(float *dest, float *a, float *b)
{
const int i = threadIdx.x;
dest[i] = a[i] * b[i];
}