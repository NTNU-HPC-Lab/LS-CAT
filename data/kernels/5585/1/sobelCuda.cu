#include "hip/hip_runtime.h"
#include "includes.h"
/*
***** sobel.cpp *****

Description:
Author:       John M. Weiss, Ph.D.
Modified By:  Jeremy Goens
Class:        CSC 461 Programming Languages
Date:         Fall 2017
Compilation   g++ -std=c++11 -g *.cpp
*/

using namespace std;

typedef unsigned char byte;

unsigned greyScale(char* str, byte*& image, byte*& pixels, unsigned int& width, unsigned int& height);
unsigned writeImage(byte*& image, unsigned int& width, unsigned int& height, char* str);
void sobel(byte*& image, byte*& edged, unsigned int& width, unsigned int& height);
void sobelOpenMP(byte*& image, byte*& edged, unsigned int& width, unsigned int& height);


/*
Description:
Author:       John M. Weiss, Ph.D.
Modified By:  Jeremy Goens
Class:        CSC 461 Programming Languages
Date:         Fall 2017
Compilation   g++ *.cpp
*/
__global__ void sobelCuda(byte* image, byte* edged, int width, int height){
int x = threadIdx.x + blockIdx.x * blockDim.x;

int j = x/width;
int i = x%width;

if( i < 1 || i >= (width-1) || j < 1 || j >= (height-1) )
return;

int gX = (-1)*image[(i-1)+((j-1)*width)];
gX += (-2)*image[(i)+((j-1)*width)];
gX += (-1)*image[(i+1)+((j-1)*width)];
gX += 1*image[(i-1)+((j+1)*width)];
gX += 2*image[(i)+((j+1)*width)];
gX += 1*image[(i+1)+((j+1)*width)];

int gY = (-1)*image[(i-1)+((j-1)*width)];
gY += 1*image[(i+1)+((j-1)*width)];
gY += (-2)*image[(i-1)+((j)*width)];
gY += 2*image[(i+1)+((j)*width)];
gY += (-1)*image[(i-1)+((j+1)*width)];
gY += 1*image[(i+1)+((j+1)*width)];

edged[i+(j*width)] = ( byte )min( sqrt( (float) (gX*gX)+(gY*gY)), 255.0);

//Black Edges all around
}