#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaPreTTGammaKernel(double *tipVector, double *l, double *r, double *umpX1, double *umpX2)
{
__shared__ volatile double ump[64];
const int tid = threadIdx.y * 4 + threadIdx.x;
if (blockIdx.y == 0)
{
ump[tid] = tipVector[4 * blockIdx.x + threadIdx.x] * l[tid];
__syncthreads();
if (threadIdx.x <= 1)
{
ump[tid] += ump[tid + 2];
}
__syncthreads();
if (threadIdx.x == 0)
{
ump[tid] += ump[tid + 1];
umpX1[blockIdx.x * 16 + threadIdx.y] = ump[tid];
}
}
else
{
ump[tid] = tipVector[4 * blockIdx.x + threadIdx.x] * r[tid];
__syncthreads();
if (threadIdx.x <= 1)
{
ump[tid] += ump[tid + 2];
}
__syncthreads();
if (threadIdx.x == 0)
{
ump[tid] += ump[tid + 1];
umpX2[blockIdx.x * 16 + threadIdx.y] = ump[tid];
}
}
}