#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaSumTIGammaKernel(unsigned char *tipX1, double *x2, double *tipVector, double *sumtable, int limit) {
const int n = blockIdx.x * blockDim.x + threadIdx.x;
if (n >= limit) {
return;
}
const int i = n / 4, l = n % 4;
double *left = &(tipVector[4 * tipX1[i]]);
double *right = &(x2[16 * i + l * 4]);
double *sum = &sumtable[i * 16 + l * 4];
#pragma unroll
for (int k = 0; k < 4; k++) {
sum[k] = left[k] * right[k];
}
}