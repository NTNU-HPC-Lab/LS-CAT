#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaEvaluateRightGammaKernel(int *wptr, double *x1, double *x2, double *diagptable, double *output, const int limit) {
const int i = blockIdx.x * blockDim.x + threadIdx.x;
output[i] = 0.0;
if (i >= limit) {
return;
}
int j;
double term = 0.0;
x1 += 16 * i;
x2 += 16 * i;
#pragma unroll
for (j = 0; j < 4; j++) {
term += x1[0] * x2[0] * diagptable[0];
term += x1[1] * x2[1] * diagptable[1];
term += x1[2] * x2[2] * diagptable[2];
term += x1[3] * x2[3] * diagptable[3];
x1 += 4;
x2 += 4;
diagptable += 4;
}
term = log(0.25 * fabs(term));
output[i] += wptr[i] * term;
}