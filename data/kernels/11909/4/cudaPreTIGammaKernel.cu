#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaPreTIGammaKernel(double *tipVector, double *l, double *ump)
{
__shared__ volatile double sump[64];
const int tid = threadIdx.y * 4 + threadIdx.x;
sump[tid] = tipVector[4 * blockIdx.x + threadIdx.x] * l[tid];
__syncthreads();
if (threadIdx.x <= 1)
{
sump[tid] += sump[tid + 2];
}
__syncthreads();
if (threadIdx.x == 0)
{
sump[tid] += sump[tid + 1];
ump[blockIdx.x * 16 + threadIdx.y] = sump[tid];
}
}