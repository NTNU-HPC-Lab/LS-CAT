#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaTIGammaKernel(double *extEV, double *x2, double *x3, unsigned char *tipX1, unsigned char *tipX2, double *r, double *uX1, double *uX2) {
__shared__ volatile double ump[64], x1px2[16], v[64];
const int tid = (threadIdx.z * 16) + (threadIdx.y * 4) + threadIdx.x;
const int offset = 16 * blockIdx.x + threadIdx.z * 4;
const int squareId = threadIdx.z * 4 + threadIdx.y;
uX1 += 16 * tipX1[blockIdx.x];
ump[tid] = x2[offset + threadIdx.x] * r[tid];
__syncthreads();
if (threadIdx.x <= 1) {
ump[tid] += ump[tid + 2];
}
__syncthreads();
if (threadIdx.x == 0) {
ump[tid] += ump[tid + 1];
uX2[4 * blockIdx.x + threadIdx.y] = ump[tid];
x1px2[squareId] = uX1[squareId] * ump[tid];
}
__syncthreads();
v[tid] = x1px2[squareId] * extEV[threadIdx.y * 4 + threadIdx.x];
__syncthreads();
if (threadIdx.y <= 1) {
v[tid] += v[tid + 8];
}
__syncthreads();
if (threadIdx.y == 0) {
v[tid] += v[tid + 4];
x3[offset + threadIdx.x] = v[tid];
}
}