#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cudaTIGammaKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *extEV = NULL;
hipMalloc(&extEV, XSIZE*YSIZE);
double *x2 = NULL;
hipMalloc(&x2, XSIZE*YSIZE);
double *x3 = NULL;
hipMalloc(&x3, XSIZE*YSIZE);
unsigned char *tipX1 = NULL;
hipMalloc(&tipX1, XSIZE*YSIZE);
unsigned char *tipX2 = NULL;
hipMalloc(&tipX2, XSIZE*YSIZE);
double *r = NULL;
hipMalloc(&r, XSIZE*YSIZE);
double *uX1 = NULL;
hipMalloc(&uX1, XSIZE*YSIZE);
double *uX2 = NULL;
hipMalloc(&uX2, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cudaTIGammaKernel<<<gridBlock,threadBlock>>>(extEV,x2,x3,tipX1,tipX2,r,uX1,uX2);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cudaTIGammaKernel<<<gridBlock,threadBlock>>>(extEV,x2,x3,tipX1,tipX2,r,uX1,uX2);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cudaTIGammaKernel<<<gridBlock,threadBlock>>>(extEV,x2,x3,tipX1,tipX2,r,uX1,uX2);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}