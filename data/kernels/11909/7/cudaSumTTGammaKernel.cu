#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaSumTTGammaKernel(unsigned char *tipX1, unsigned char *tipX2, double *tipVector, double *sumtable, int limit) {
const int n = blockIdx.x * blockDim.x + threadIdx.x;
if (n >= limit) {
return;
}
const int i = n / 4, j = n % 4;
double *left = &(tipVector[4 * tipX1[i]]);
double *right = &(tipVector[4 * tipX2[i]]);
double *sum = &sumtable[i * 16 + j * 4];
#pragma unroll
for (int k = 0; k < 4; k++) {
sum[k] = left[k] * right[k];
}
}