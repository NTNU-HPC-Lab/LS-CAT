#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaSumIIGammaKernel(double *x1, double *x2, double *sumtable, int limit) {
const int n = blockIdx.x * blockDim.x + threadIdx.x;
if (n >= limit) {
return;
}
const int i = n / 4, l = n % 4;
double *left = &(x1[16 * i + l * 4]);
double *right = &(x2[16 * i + l * 4]);
double *sum = &(sumtable[i * 16 + l * 4]);
#pragma unroll
for (int k = 0; k < 4; k++) {
sum[k] = left[k] * right[k];
}
}