#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaIIGammaKernel(double *extEV, double *x1, double *x2, double *x3, double *left, double *right) {
__shared__ volatile double al[64], ar[64], v[64], x1px2[16];
const int tid = (threadIdx.z * 16) + (threadIdx.y * 4) + threadIdx.x;
const int offset = 16 * blockIdx.x + 4 * threadIdx.z;
al[tid] = x1[offset + threadIdx.x] * left[tid];
ar[tid] = x2[offset + threadIdx.x] * right[tid];
__syncthreads();
if (threadIdx.x <= 1) {
al[tid] += al[tid + 2];
ar[tid] += ar[tid + 2];
}
__syncthreads();
if (threadIdx.x == 0) {
al[tid] += al[tid + 1];
ar[tid] += ar[tid + 1];
x1px2[(threadIdx.z * 4) + threadIdx.y] = al[tid] * ar[tid];
}
__syncthreads();
v[tid] = x1px2[threadIdx.y + (threadIdx.z * 4)] *
extEV[threadIdx.y * 4 + threadIdx.x];
__syncthreads();
if (threadIdx.y <= 1) {
v[tid] += v[tid + 8];
}
__syncthreads();
if (threadIdx.y == 0) {
v[tid] += v[tid + 4];
x3[offset + threadIdx.x] = v[tid];
}
}