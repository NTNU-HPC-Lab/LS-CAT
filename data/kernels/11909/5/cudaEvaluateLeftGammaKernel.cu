#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaEvaluateLeftGammaKernel(int *wptr, double *x2, double *tipVector, unsigned char *tipX1, double *diagptable, double *output, const int limit) {
const int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= limit) {
output[i] = 0.0;
return;
}
int j;
double term = 0.0;
tipVector += 4 * tipX1[i];
x2 += 16 * i;
#pragma unroll
for (j = 0; j < 4; j++) {
term += tipVector[0] * x2[0] * diagptable[0];
term += tipVector[1] * x2[1] * diagptable[1];
term += tipVector[2] * x2[2] * diagptable[2];
term += tipVector[3] * x2[3] * diagptable[3];
x2 += 4;
diagptable += 4;
}
term = log(0.25 * fabs(term));
output[i] = wptr[i] * term;
}