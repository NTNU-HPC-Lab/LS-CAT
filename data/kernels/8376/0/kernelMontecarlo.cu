#include "hip/hip_runtime.h"
#include "includes.h"
#define N 128*128



__global__ void kernelMontecarlo(float *x, float *y,int *contador) {
//int i = threadIdx.x + blockIdx.x*blockDim.x;
//int j = threadIdx.y + blockIdx.y*blockDim.y;
int indice = threadIdx.x + blockIdx.x*blockDim.x;
//int indice=i;
//printf("Indice: %f\n",(x[indice]*x[indice] + y[indice]*y[indice]));
if((x[indice]*x[indice] + y[indice]*y[indice]) <=1.0) {
atomicAdd(contador,1);//contador++;
//printf("Contador: %d\n",*contador);
}
}