#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMultMatrices(float *a, float *b, float *c,int m, int n) {
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;
//printf("%d,%d\n",i,j);
c[j+i*n]=0;
for(int k=0;k<N;k++) c[j+i*n]+=a[j+k*n]*b[k+i*n];;
__syncthreads();
}