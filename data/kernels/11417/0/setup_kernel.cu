#include "hip/hip_runtime.h"
#include "includes.h"



#define PI 3.14159265359

#define DEG_TO_RAD (PI / 180.0)

typedef unsigned char byte;

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
hiprand_init(seed, id, 0, &state[id]);
}