#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputePhiMag_GPU(float* phiR, float* phiI, float* phiMag, int numK) {
int indexK = blockIdx.x*KERNEL_PHI_MAG_THREADS_PER_BLOCK + threadIdx.x;
if (indexK < numK) {
float real = phiR[indexK];
float imag = phiI[indexK];
phiMag[indexK] = real*real + imag*imag;
}
}