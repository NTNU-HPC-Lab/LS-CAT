#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(unsigned int n, unsigned int *data, unsigned int *inter)
{

__shared__ unsigned int uni;
if (threadIdx.x == 0) { uni = inter[blockIdx.x]; }
__syncthreads();

unsigned int g_ai = blockIdx.x*2*blockDim.x + threadIdx.x;
unsigned int g_bi = g_ai + blockDim.x;

if (g_ai < n) { data[g_ai] += uni; }
if (g_bi < n) { data[g_bi] += uni; }
}