#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void splitRearrange (int numElems, int iter, unsigned int* keys_i, unsigned int* keys_o, unsigned int* values_i, unsigned int* values_o, unsigned int* histo){
__shared__ unsigned int histo_s[(1<<BITS)];
__shared__ unsigned int array_s[4*SORT_BS];
int index = blockIdx.x*4*SORT_BS + 4*threadIdx.x;

if (threadIdx.x < (1<<BITS)){
histo_s[threadIdx.x] = histo[gridDim.x*threadIdx.x+blockIdx.x];
}

uint4 mine, value;
if (index < numElems){
mine = *((uint4*)(keys_i+index));
value = *((uint4*)(values_i+index));
} else {
mine.x = UINT32_MAX;
mine.y = UINT32_MAX;
mine.z = UINT32_MAX;
mine.w = UINT32_MAX;
}
uint4 masks = {(mine.x&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
(mine.y&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
(mine.z&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
(mine.w&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)};

((uint4*)array_s)[threadIdx.x] = masks;
__syncthreads();

uint4 new_index = {histo_s[masks.x],histo_s[masks.y],histo_s[masks.z],histo_s[masks.w]};

int i = 4*threadIdx.x-1;
while (i >= 0){
if (array_s[i] == masks.x){
new_index.x++;
i--;
} else {
break;
}
}

new_index.y = (masks.y == masks.x) ? new_index.x+1 : new_index.y;
new_index.z = (masks.z == masks.y) ? new_index.y+1 : new_index.z;
new_index.w = (masks.w == masks.z) ? new_index.z+1 : new_index.w;

if (index < numElems){
keys_o[new_index.x] = mine.x;
values_o[new_index.x] = value.x;

keys_o[new_index.y] = mine.y;
values_o[new_index.y] = value.y;

keys_o[new_index.z] = mine.z;
values_o[new_index.z] = value.z;

keys_o[new_index.w] = mine.w;
values_o[new_index.w] = value.w;
}
}