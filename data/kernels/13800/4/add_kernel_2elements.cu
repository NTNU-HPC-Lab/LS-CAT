#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel_2elements(int* device_result, int* device_blocksum_2elements)
{
__shared__ int temp1;
int thid = threadIdx.x;
int N = blockDim.x;
if (thid == 0) temp1 = device_blocksum_2elements[blockIdx.x];
__syncthreads();
device_result[blockIdx.x * 4 * blockDim.x + thid] = device_result[blockIdx.x * 4 * blockDim.x + thid] + temp1;
device_result[blockIdx.x * 4 * blockDim.x + thid + N] =
device_result[blockIdx.x * 4 * blockDim.x + thid + N] + temp1;
device_result[blockIdx.x * 4 * blockDim.x + thid + 2 * N] =
device_result[blockIdx.x * 4 * blockDim.x + thid + 2 * N] + temp1;
device_result[blockIdx.x * 4 * blockDim.x + thid + 3 * N] =
device_result[blockIdx.x * 4 * blockDim.x + thid + 3 * N] + temp1;
}