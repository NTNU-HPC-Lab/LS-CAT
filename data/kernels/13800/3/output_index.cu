#include "hip/hip_runtime.h"
#include "includes.h"



#define ITER 4
#define BANK_OFFSET1(n) (n) + (((n) >> 5))
#define BANK_OFFSET(n) (n) + (((n) >> 5))
#define NUM_BLOCKS(length, dim) nextPow2(length) / (2 * dim)
#define ELEM 4
#define TOTAL_THREADS 512
#define TWO_PWR(n) (1 << (n))
extern float toBW(int bytes, float sec);

__device__ __inline__ void prefix_sum_warp(int thid, int* temp, int N)
{
if (thid < 16)
{
int i = temp[thid];
if (thid >= 1) temp[thid] += temp[thid - 1];
if (thid >= 2) temp[thid] += temp[thid - 2];
if (thid >= 4) temp[thid] += temp[thid - 4];
if (thid >= 8) temp[thid] += temp[thid - 8];
temp[thid] -= i;
}
}
__global__ void output_index(int* device_hist, int* pdevice_hist, int* device_input, int* device_out, int length, int num_blocks, int nibble)
{
__shared__ int temp[TWO_PWR(ITER)];
int t = 4 * blockIdx.x * blockDim.x + threadIdx.x;
int N = TOTAL_THREADS;
int thid = threadIdx.x;

if (t < length)
{
int val1;
int val2;
int val3;
int val4;
int nibble1 = nibble << 2;
int lindex1;
int lindex2;
int lindex3;
int lindex4;
int gindex1;
int gindex2;
int gindex3;
int gindex4;
int a = t;
int b = t + 1 * N;
int c = t + 2 * N;
int d = t + 3 * N;
int a1 = thid;
int b1 = thid + 1 * N;
int c1 = thid + 2 * N;
int d1 = thid + 3 * N;
val1 = device_input[a];
val2 = device_input[b];
val3 = device_input[c];
val4 = device_input[d];

if (thid < 32)
{
if ((thid) < ITER)
{
temp[4 * thid] = device_hist[4 * thid * num_blocks + blockIdx.x];
temp[4 * thid + 1] = device_hist[(4 * thid + 1) * num_blocks + blockIdx.x];
temp[4 * thid + 2] = device_hist[(4 * thid + 2) * num_blocks + blockIdx.x];
temp[4 * thid + 3] = device_hist[(4 * thid + 3) * num_blocks + blockIdx.x];
}

prefix_sum_warp(thid, temp, TWO_PWR(ITER));
}
__syncthreads();
lindex1 = temp[((val1 >> (nibble1)) & ((1 << ITER) - 1))];
lindex2 = temp[((val2 >> (nibble1)) & ((1 << ITER) - 1))];
lindex3 = temp[((val3 >> (nibble1)) & ((1 << ITER) - 1))];
lindex4 = temp[((val4 >> (nibble1)) & ((1 << ITER) - 1))];
gindex1 = pdevice_hist[((val1 >> (nibble1)) & ((1 << ITER) - 1)) * num_blocks + blockIdx.x];
gindex2 = pdevice_hist[((val2 >> (nibble1)) & ((1 << ITER) - 1)) * num_blocks + blockIdx.x];
gindex3 = pdevice_hist[((val3 >> (nibble1)) & ((1 << ITER) - 1)) * num_blocks + blockIdx.x];
gindex4 = pdevice_hist[((val4 >> (nibble1)) & ((1 << ITER) - 1)) * num_blocks + blockIdx.x];

device_out[a1 + gindex1 - lindex1] = val1;
device_out[b1 + gindex2 - lindex2] = val2;
device_out[c1 + gindex3 - lindex3] = val3;
device_out[d1 + gindex4 - lindex4] = val4;
}
}