#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemmNT( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
// Partial results
float c[TILE_N];
for (int i=0; i < TILE_N; i++)
c[i] = 0.0f;
int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
int m = blockIdx.x * TILE_M + mid;
int n = blockIdx.y * TILE_N + threadIdx.x;
__shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
for (int i = 0; i < k; i+=TILE_TB_HEIGHT) {
float a;
b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*ldb];
__syncthreads();
for (int j = 0; j < TILE_TB_HEIGHT; j++) {
a = A[m + (i+j)*lda];
for (int kk = 0; kk < TILE_N; kk++)
c[kk] += a * b_s[j][kk];

}
__syncthreads();
}
int t = ldc*blockIdx.y * TILE_N + m;
for (int i = 0; i < TILE_N; i++) {
C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
}
}