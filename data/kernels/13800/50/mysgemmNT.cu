#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemmNT( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
float c = 0.0f;
int m = blockIdx.x * blockDim.x + threadIdx.x;
int n = blockIdx.y * blockDim.y + threadIdx.y;
for (int i = 0; i < k; ++i) {
float a = A[m + i * lda];
float b = B[n + i * ldb];
c += a * b;
}
C[m+n*ldc] = C[m+n*ldc] * beta + alpha * c;
}