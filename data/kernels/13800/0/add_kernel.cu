#include "hip/hip_runtime.h"
#include "includes.h"



#define ITER 4
#define BANK_OFFSET1(n) (n) + (((n) >> 5))
#define BANK_OFFSET(n) (n) + (((n) >> 5))
#define NUM_BLOCKS(length, dim) nextPow2(length) / (2 * dim)
#define ELEM 4
#define TOTAL_THREADS 512
#define TWO_PWR(n) (1 << (n))
extern float toBW(int bytes, float sec);

__global__ void add_kernel(int* device_result, int* device_blocksum)
{
int temp1;
int thid = threadIdx.x;
int N = blockDim.x;
int offset = blockIdx.x * 4 * blockDim.x;

temp1 = device_blocksum[blockIdx.x];
device_result[offset + thid] = device_result[offset + thid] + temp1;
device_result[offset + thid + N] = device_result[offset + thid + N] + temp1;
device_result[offset + thid + 2 * N] = device_result[offset + thid + 2 * N] + temp1;
device_result[offset + thid + 3 * N] = device_result[offset + thid + 3 * N] + temp1;
}