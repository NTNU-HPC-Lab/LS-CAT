#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "calculate_sumterm_part.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double2 *Up = NULL;
hipMalloc(&Up, XSIZE*YSIZE);
double2 *Vpl = NULL;
hipMalloc(&Vpl, XSIZE*YSIZE);
const double2 *A_t = NULL;
hipMalloc(&A_t, XSIZE*YSIZE);
const double *SR = NULL;
hipMalloc(&SR, XSIZE*YSIZE);
const unsigned char *nonzero_midx1234s = NULL;
hipMalloc(&nonzero_midx1234s, XSIZE*YSIZE);
const unsigned int N = 1;
const unsigned int M = 1;
const double SK_factor = 1;
const unsigned int NUM_NONZERO = 1;
const unsigned int NUM_MODES = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_sumterm_part<<<gridBlock,threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
calculate_sumterm_part<<<gridBlock,threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
calculate_sumterm_part<<<gridBlock,threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}