#include "hip/hip_runtime.h"
#include "includes.h"
/***
* File: maxwell_griffin_lab4p2.cu
* Desc: Performs 2 Sobel edge detection operations on a .bmp, once by a
*       serial algorithm, and once by a massively parallel CUDA algorithm.
*/


extern "C"
{
}

#define PIXEL_BLACK (0)
#define PIXEL_WHITE (255)
#define PERCENT_BLACK_THRESHOLD (0.75)

#define CUDA_GRIDS (1)
#define CUDA_BLOCKS_PER_GRID (32)
#define CUDA_THREADS_PER_BLOCK (128)

#define MS_PER_SEC (1000)
#define NS_PER_MS (1000 * 1000)
#define NS_PER_SEC (NS_PER_MS * MS_PER_SEC)

#define LINEARIZE(row, col, dim) \
(((row) * (dim)) + (col))

static struct timespec rtcSerialStart;
static struct timespec rtcSerialEnd;
static struct timespec rtcParallelStart;
static struct timespec rtcParallelEnd;

__device__ int Sobel_Gx[3][3] = {
{ -1, 0, 1 },
{ -2, 0, 2 },
{ -1, 0, 1 }
};

__device__ int Sobel_Gy[3][3] = {
{  1,  2,  1 },
{  0,  0,  0 },
{ -1, -2, -1 }
};

/*
* Display all header information and matrix and CUDA parameters.
*
* @param inputFile -- name of the input image
* @param serialOutputFile -- name of the serial output image
* @param parallelOutputFile -- name of the parallel output image
* @param imageHeight -- in pixels
* @param imageWidth -- in pixels
*/
void DisplayParameters(
char *inputFile,
char *serialOutputFile,
char *cudaOutputFile,
int imageHeight,
int imageWidth)
{
printf("********************************************************************************\n");
printf("lab4p2: serial vs. CUDA Sobel edge detection.\n");
printf("\n");
printf("Input image: %s \t(Height: %d pixels, width: %d pixels)\n", inputFile, imageHeight, imageWidth);
printf("Serial output image: \t%s\n", serialOutputFile);
printf("CUDA output image: \t%s\n", cudaOutputFile);
printf("\n");
printf("CUDA compute structure:\n");
printf("|-- with %d grid\n", CUDA_GRIDS);
printf("    |-- with %d blocks\n", CUDA_BLOCKS_PER_GRID);
printf("        |-- with %d threads per block\n", CUDA_THREADS_PER_BLOCK);
printf("\n");
}

/*
* Display the timing and convergence results to the screen.
*
* @param serialConvergenceThreshold
* @param serialConvergenceThreshold
*/
void DisplayResults(
int serialConvergenceThreshold,
int parallelConvergenceThreshold)
{
printf("Time taken for serial Sobel edge detection: %lf\n",
(LINEARIZE(rtcSerialEnd.tv_sec, rtcSerialEnd.tv_nsec, NS_PER_SEC)
- LINEARIZE(rtcSerialStart.tv_sec, rtcSerialStart.tv_nsec, NS_PER_SEC))
/ ((double)NS_PER_SEC));

printf("Convergence Threshold: %d\n", serialConvergenceThreshold);
printf("\n");

printf("Time taken for CUDA Sobel edge detection: %lf\n",
(LINEARIZE(rtcParallelEnd.tv_sec, rtcParallelEnd.tv_nsec, NS_PER_SEC)
- LINEARIZE(rtcParallelStart.tv_sec, rtcParallelStart.tv_nsec, NS_PER_SEC))
/ ((double)NS_PER_SEC));

printf("Convergence Threshold: %d\n", parallelConvergenceThreshold);
printf("********************************************************************************\n");
}

/*
* Serial algorithm to keep perform a Sobel edge detection on an input pixel
* buffer at different brightness thresholds until a certain percentage of
* pixels in the output pixel buffer are black.
*
* @param input -- input pixel buffer
* @param output -- output pixel buffer
* @param height -- height of pixel image
* @param width -- width of pixel image
* @return -- gradient threshold at which PERCENT_BLACK_THRESHOLD pixels are black
*/
__global__ void CudaSobelEdgeDetection(uint8_t *input, uint8_t *output, int height, int width, int gradientThreshold)
{
int row = 0;
for(int i = 0; row < (height - 1); i++)
{
// Let the blockIdx increment beyond its dimension for cyclic distribution of the test pixels
int blockRow = (i * gridDim.x) + blockIdx.x;

// Calculate the row/col in the image buffer that this thread's stencil's center is on
row = (LINEARIZE(blockRow, threadIdx.x, blockDim.x) / (width - 2)) + 1;
int col = (LINEARIZE(blockRow, threadIdx.x, blockDim.x) % (width - 2)) + 1;

// Calculate Sobel magnitude of gradient directly, instead of using Sobel_Magnitude utility
double Gx = (Sobel_Gx[0][0] * input[LINEARIZE(row - 1, col - 1, width)])
+ (Sobel_Gx[0][2] * input[LINEARIZE(row - 1, col + 1, width)])
+ (Sobel_Gx[1][0] * input[LINEARIZE(row, col - 1, width)])
+ (Sobel_Gx[1][2] * input[LINEARIZE(row, col + 1, width)])
+ (Sobel_Gx[2][0] * input[LINEARIZE(row + 1, col - 1, width)])
+ (Sobel_Gx[2][2] * input[LINEARIZE(row + 1, col + 1, width)]);

double Gy = (Sobel_Gy[0][0] * input[LINEARIZE(row - 1, col - 1, width)])
+ (Sobel_Gy[0][1] * input[LINEARIZE(row - 1, col, width)])
+ (Sobel_Gy[0][2] * input[LINEARIZE(row - 1, col + 1, width)])
+ (Sobel_Gy[2][0] * input[LINEARIZE(row + 1, col - 1, width)])
+ (Sobel_Gy[2][1] * input[LINEARIZE(row + 1, col, width)])
+ (Sobel_Gy[2][2] * input[LINEARIZE(row + 1, col + 1, width)]);

if(((Gx * Gx) + (Gy * Gy)) > (gradientThreshold * gradientThreshold))
{
output[LINEARIZE(row, col, width)] = PIXEL_WHITE;
}
else
{
output[LINEARIZE(row, col, width)] = PIXEL_BLACK;
}
}
}