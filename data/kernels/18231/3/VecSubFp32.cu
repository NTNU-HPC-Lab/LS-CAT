#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecSubFp32(float* in0, float* in1, float* out, int cnt)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < cnt) {
out[tid] = in0[tid] - in1[tid];
}
}