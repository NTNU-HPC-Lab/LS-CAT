#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_reduc(int* data, int* len, int* width){
int indx = blockIdx.x * gridDim.x + threadIdx.x;
int sum = 0;
for (int i=indx; i<indx + *width; i++){
if (i < *len)
sum += data[i];
}
data[indx] = sum;
}