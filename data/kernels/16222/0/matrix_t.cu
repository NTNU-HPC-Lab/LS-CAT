#include "hip/hip_runtime.h"
#include "includes.h"


/*
WRITE CUDA KERNEL FOR TRANSPOSE HERE
*/
const int CHUNK_SIZE = 32;
const int CHUNK_ROWS = 8;


__global__ void matrix_t(int* data, int* out, int* rows, int* cols){
__shared__ int chunk[CHUNK_SIZE][CHUNK_SIZE];
int x = blockIdx.x * CHUNK_SIZE + threadIdx.x;
int y = blockIdx.y * CHUNK_SIZE + threadIdx.y;

for (int i=0; i<CHUNK_SIZE; i+= CHUNK_ROWS) {
chunk[threadIdx.x][threadIdx.y+i] = data[(y + i) * *cols + x];
}
__syncthreads();

x = blockIdx.y * CHUNK_SIZE + threadIdx.x;
y = blockIdx.x * CHUNK_SIZE + threadIdx.y;

for (int i=0; i<CHUNK_SIZE; i+= CHUNK_ROWS) {
if (x < *rows && y+i < *cols) {
out[(y + i) * *rows + x] = chunk[threadIdx.y + i][threadIdx.x];
//            out[(y + i) * *rows + x] = 1;
}
}
}