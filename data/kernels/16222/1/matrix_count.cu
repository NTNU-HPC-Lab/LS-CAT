#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_count(int* data, int* count, int* rows, int* cols){
__shared__ int chunk[CHUNK_SIZE][CHUNK_SIZE];
int x = blockIdx.x * CHUNK_SIZE + threadIdx.x;
int y = blockIdx.y * CHUNK_SIZE + threadIdx.y;

for (int i=0; i<CHUNK_SIZE; i+= CHUNK_ROWS) {
chunk[threadIdx.x][threadIdx.y+i] = data[(y + i) * *cols + x];
}
__syncthreads();

x = blockIdx.y * CHUNK_SIZE + threadIdx.x;
y = blockIdx.x * CHUNK_SIZE + threadIdx.y;

for (int i=0; i<CHUNK_SIZE; i+= CHUNK_ROWS) {
if (x < *rows && y+i < *cols) {
if (chunk[threadIdx.y + i][threadIdx.x] == 1)
atomicAdd(count, 1);
}
}
}