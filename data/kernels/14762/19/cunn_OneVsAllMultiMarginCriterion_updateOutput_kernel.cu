#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_OneVsAllMultiMarginCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
__shared__ float buffer[MULTIMARGIN_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *output_k = output + k;
int target_k = ((int)target[k])-1;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

buffer[threadIdx.x] = 0;
for(int i = i_start; i < i_end; i += i_step)
{

float y = (i==target_k) ? 1.0 : -1.0;
float z = 1 - input_k[i]*y;
if(z > 0){
float weight = (i==target_k) ? positiveWeight[i] : 1.0;
buffer[threadIdx.x] += z*weight;
}
}
__syncthreads();

// reduce
if (threadIdx.x == 0)
{
float sum = 0;
for (int i=0; i<blockDim.x; i++)
sum += buffer[i];

if(sizeaverage)
*output_k = sum/dim;
else
*output_k = sum;
}
}