#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
// output offset:
long o = threadIdx.x + blockDim.x * blockIdx.x;
if (o >= nrows) return;

// input offset:
long i = o * ncols;

// bprop min gradient:
long idx = indices[o]-1;
input[i+idx] = output[o];
}