#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_OneVsAllNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
__shared__ float buffer[NLL_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *output_k = output + k;
int target_k = ((int)target[k])-1;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

buffer[threadIdx.x] = 0;
for(int i = i_start; i < i_end; i += i_step)
{
float z = (i==target_k) ?  (-positiveWeight[i]*log(input_k[i])) : -log(1 - input_k[i]);
buffer[threadIdx.x] += z;
}
__syncthreads();

// reduce
if (threadIdx.x == 0)
{
float sum = 0;
for (int i=0; i<blockDim.x; i++)
sum += buffer[i];

if(sizeaverage)
*output_k = sum/nframe;
else
*output_k = sum;
}
}