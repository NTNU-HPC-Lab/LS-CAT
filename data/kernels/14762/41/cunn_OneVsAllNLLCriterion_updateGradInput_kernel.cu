#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_OneVsAllNLLCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
// __shared__ float buffer[NLL_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *gradInput_k = gradInput + k*dim;
int target_k = ((int)target[k])-1;
float g = (sizeaverage ? 1./((float)nframe) : 1.);

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

//  buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float z = (i==target_k) ? -g*positiveWeight[i]/input_k[i] : g/(1 - input_k[i]);
gradInput_k[i] = z;
}

__syncthreads();

// reduce
//if (threadIdx.x == 0)
//{
// float gradInput_target_k = 0;
//for (int i=0; i<blockDim.x; i++)
// gradInput_target_k += buffer[i];
//gradInput_k[target_k] = gradInput_target_k;
//}
}