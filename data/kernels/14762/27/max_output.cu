#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols)
{
// output offset:
long o = threadIdx.x + blockDim.x * blockIdx.x;
if (o >= nrows) return;

// input offset:
long i = o * ncols;

// move pointers
input = input + i;

// compute max:
float max = input[0];
long argmax = 0;
long ii;
for (ii=1; ii<ncols; ii++) {
float val = input[ii];
if (val > max) {
max = val;
argmax = ii;
}
}

// store
output[o] = max;
indices[o] = argmax+1;
}