#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
__shared__ float shInputs[NTHREADS];
// Verify whether `register` does anything here.
register int i, j, t;

shInputs[threadIdx.x] = .0;
for (i = threadIdx.x; i < nframe; i += NTHREADS) {
for (j = 0; j < ntarget; ++j) {
t = (int)target[i * ntarget + j] - 1;
if (t >= 0)
shInputs[threadIdx.x] += input[i * ndim + t];
}
}
__syncthreads();

// TODO: T4951791 Reuse code between updateOutput_kernel1 and
// updateOutput_kernel
if (threadIdx.x == 0) {
*output = .0;
for (i = 0; i < NTHREADS; ++i)
*output += shInputs[i];
if (sizeAverage)
*output /= nframe;
*output = -(*output);
}
}