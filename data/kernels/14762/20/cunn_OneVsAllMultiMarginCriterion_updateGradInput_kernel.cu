#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_OneVsAllMultiMarginCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
// __shared__ float buffer[MULTIMARGIN_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *gradInput_k = gradInput + k*dim;
int target_k = ((int)target[k])-1;
float g = (sizeaverage ? 1./((float)dim) : 1.);

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

//  buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float y = (i==target_k) ? 1.0 : -1.0;
float z = 1 - input_k[i]*y;

if(z > 0)
{
float weight = (i==target_k) ? positiveWeight[i] : 1.0;
float h =  -y*g*weight;
gradInput_k[i] = h;
}
else
gradInput_k[i] = 0;
}

__syncthreads();

// reduce
//if (threadIdx.x == 0)
//{
// float gradInput_target_k = 0;
//for (int i=0; i<blockDim.x; i++)
// gradInput_target_k += buffer[i];
//gradInput_k[target_k] = gradInput_target_k;
//}
}