#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
// output offset:
long o = threadIdx.x + blockDim.x * blockIdx.x;
if (o >= nrows) return;

// input offset:
long i = o * ncols;

// bprop max gradient:
long idx = indices[o]-1;
input[i+idx] = output[o];
}