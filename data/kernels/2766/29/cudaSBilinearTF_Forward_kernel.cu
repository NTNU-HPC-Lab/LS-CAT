#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSBilinearTF_Forward_kernel( unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const unsigned int* yLowIdx, const unsigned int* yHighIdx, const float* yInter, const unsigned int* xLowIdx, const unsigned int* xHighIdx, const float* xInter, const float* input, float* outputs)
{

const unsigned int inputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*inputWidth*inputHeight;

const unsigned int outputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*outputWidth*outputHeight;
for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x)
{
for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y)
{
for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x)
{
const unsigned int indexTL = xLowIdx[ox] + yLowIdx[oy]*inputWidth
+ ch*inputWidth*inputHeight
+ inputOffset;

const unsigned int indexTR = xHighIdx[ox] + yLowIdx[oy]*inputWidth
+ ch*inputWidth*inputHeight
+ inputOffset;

const unsigned int indexBL = xLowIdx[ox] + yHighIdx[oy]*inputWidth
+ ch*inputWidth*inputHeight
+ inputOffset;

const unsigned int indexBR = xHighIdx[ox] + yHighIdx[oy]*inputWidth
+ ch*inputWidth*inputHeight
+ inputOffset;

const float top_left = input[indexTL];
const float top_right = input[indexTR];
const float bottom_left = input[indexBL];
const float bottom_right = input[indexBR];

const float top = top_left + (top_right - top_left) * xInter[ox];
const float bottom = bottom_left + (bottom_right - bottom_left) * xInter[ox];

outputs[ ox + oy*outputWidth
+ ch*outputWidth*outputHeight + outputOffset]  = top + (bottom - top) * yInter[oy];

}
}
}
}