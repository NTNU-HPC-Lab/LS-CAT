#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSMult_kernel(unsigned int size, float* a, float* b, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride)
result[i] = a[i] * b[i] + beta * result[i];
}
else {
for (unsigned int i = index; i < size; i += stride)
result[i] = a[i] * b[i];
}
}