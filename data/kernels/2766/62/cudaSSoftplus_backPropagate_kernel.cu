#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSoftplus_backPropagate_kernel(float* x, float* dx, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
dx[i] *= (1.0f - exp(-x[i]));
}
}