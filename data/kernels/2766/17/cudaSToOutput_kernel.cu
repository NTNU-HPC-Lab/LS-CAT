#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSToOutput_kernel( const unsigned int nbProposals, const unsigned int scoreIdx, const unsigned int nbCls, const unsigned int nbOutputs, const unsigned int maxParts, const unsigned int maxTemplates, bool generateParts, bool generateTemplates, const int* numPartsPerClass, const int* numTemplatesPerClass, const int* maxCls, const float* ROIEst, const int* predictionIndex, const float* partsPrediction, const float* partsVisibilityPrediction, const float* templatesPrediction, float* outputs)
{
const int batchPos = blockIdx.z*nbProposals;
const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

if(index < nbProposals)
{
const unsigned int inputIdx = index*4*(nbCls - scoreIdx)
+ batchPos*4*(nbCls - scoreIdx);
unsigned int outputIdx = 0;
unsigned offset = 0;

if((nbOutputs == 4))
outputIdx = index*4 + batchPos*4;
else if((nbOutputs == 5))
outputIdx = index*5 + batchPos*5;
else if(generateParts && generateTemplates)
outputIdx = (index + batchPos)*(5 + maxParts*3 + maxTemplates*3);
else if(generateTemplates)
outputIdx = (index + batchPos)*(5 + maxTemplates*3);
else if(generateParts)
outputIdx = (index + batchPos)*(5 + maxParts*3);

outputs[0 + outputIdx] = ROIEst[0 + inputIdx];
outputs[1 + outputIdx] = ROIEst[1 + inputIdx];
outputs[2 + outputIdx] = ROIEst[2 + inputIdx];
outputs[3 + outputIdx] = ROIEst[3 + inputIdx];

offset += 4;

if(nbOutputs > 4)
{
int cls = maxCls[index + batchPos];
outputs[4 + outputIdx] = cls > -1 ?
(float) cls
: 0.0;
offset += 1;
}

if(generateParts)
{
const int predProp = predictionIndex[(index + batchPos)*2 + 0];
const int predCls = predictionIndex[(index + batchPos)*2 + 1];

if(predCls > -1)
{
// PARTS PROCESSING
for(unsigned int part = 0; part < numPartsPerClass[predCls];
++part)
{
const unsigned int partIdx = batchPos*maxParts*2*nbCls
+ predProp*maxParts*2*nbCls
+ predCls*maxParts*2
+ part*2;
outputs[0 + offset + part*2 + outputIdx] = partsPrediction[0 + partIdx];
outputs[1 + offset + part*2 + outputIdx] = partsPrediction[1 + partIdx];

}
for(int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx)
{
outputs[0 + offset + numPartsPerClass[predCls]*2 + idx*2 + outputIdx] = 0.0;
outputs[1 + offset + numPartsPerClass[predCls]*2 + idx*2 + outputIdx] = 0.0;
}
}

offset += 2*maxParts;

if(predCls > -1)
{
// PARTS VISIBILITY PROCESSING
for(unsigned int part = 0; part < numPartsPerClass[predCls];
++part)
{
const unsigned int partVisibilityIdx = batchPos*maxParts*nbCls
+ predProp*maxParts*nbCls
+ predCls*maxParts
+ part;
outputs[offset + part + outputIdx] = partsVisibilityPrediction[partVisibilityIdx];

}

for(int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx)
outputs[offset + numPartsPerClass[predCls] + idx + outputIdx] = -1.0;
}
offset += maxParts;
}

if(generateTemplates)
{

const int predProp = predictionIndex[(index + batchPos)*2 + 0];
const int predCls = predictionIndex[(index + batchPos)*2 + 1];

if(predCls > -1)
{
for(unsigned int tpl = 0; tpl < numTemplatesPerClass[predCls]; ++tpl)
{
unsigned int templateIdx = batchPos*maxTemplates*3*nbCls
+ predProp*maxTemplates*3*nbCls
+ predCls*maxTemplates*3
+ tpl*3;

outputs[0 + offset + tpl*3 + outputIdx] = templatesPrediction[0 + templateIdx];
outputs[1 + offset + tpl*3 + outputIdx] = templatesPrediction[1 + templateIdx];
outputs[2 + offset + tpl*3 + outputIdx] = templatesPrediction[2 + templateIdx];

}
for(int idx = numTemplatesPerClass[predCls]; idx < maxParts; ++idx)
{
outputs[0 + offset + numTemplatesPerClass[predCls]*3 + idx*3 + outputIdx] = 0.0;
outputs[1 + offset + numTemplatesPerClass[predCls]*3 + idx*3 + outputIdx] = 0.0;
outputs[2 + offset + numTemplatesPerClass[predCls]*3 + idx*3 + outputIdx] = 0.0;

}

}
}
}

}