#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cudaSToOutput_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const unsigned int nbProposals = 1;
const unsigned int scoreIdx = 1;
const unsigned int nbCls = 1;
const unsigned int nbOutputs = 1;
const unsigned int maxParts = 1;
const unsigned int maxTemplates = 1;
bool generateParts = 1;
bool generateTemplates = 1;
const int *numPartsPerClass = NULL;
hipMalloc(&numPartsPerClass, XSIZE*YSIZE);
const int *numTemplatesPerClass = NULL;
hipMalloc(&numTemplatesPerClass, XSIZE*YSIZE);
const int *maxCls = NULL;
hipMalloc(&maxCls, XSIZE*YSIZE);
const float *ROIEst = NULL;
hipMalloc(&ROIEst, XSIZE*YSIZE);
const int *predictionIndex = NULL;
hipMalloc(&predictionIndex, XSIZE*YSIZE);
const float *partsPrediction = NULL;
hipMalloc(&partsPrediction, XSIZE*YSIZE);
const float *partsVisibilityPrediction = NULL;
hipMalloc(&partsVisibilityPrediction, XSIZE*YSIZE);
const float *templatesPrediction = NULL;
hipMalloc(&templatesPrediction, XSIZE*YSIZE);
float *outputs = NULL;
hipMalloc(&outputs, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cudaSToOutput_kernel<<<gridBlock,threadBlock>>>(nbProposals,scoreIdx,nbCls,nbOutputs,maxParts,maxTemplates,generateParts,generateTemplates,numPartsPerClass,numTemplatesPerClass,maxCls,ROIEst,predictionIndex,partsPrediction,partsVisibilityPrediction,templatesPrediction,outputs);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cudaSToOutput_kernel<<<gridBlock,threadBlock>>>(nbProposals,scoreIdx,nbCls,nbOutputs,maxParts,maxTemplates,generateParts,generateTemplates,numPartsPerClass,numTemplatesPerClass,maxCls,ROIEst,predictionIndex,partsPrediction,partsVisibilityPrediction,templatesPrediction,outputs);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cudaSToOutput_kernel<<<gridBlock,threadBlock>>>(nbProposals,scoreIdx,nbCls,nbOutputs,maxParts,maxTemplates,generateParts,generateTemplates,numPartsPerClass,numTemplatesPerClass,maxCls,ROIEst,predictionIndex,partsPrediction,partsVisibilityPrediction,templatesPrediction,outputs);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}