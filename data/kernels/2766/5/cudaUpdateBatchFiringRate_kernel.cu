#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateBatchFiringRate_kernel(unsigned int * firingRate, unsigned int * batchFiringRate, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ, unsigned int batchSize)
{

const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;

for (unsigned int channel = blockIdx.x; channel < inputsDimZ; channel += gridDim.x){
for (unsigned int sy = 0; sy < inputsDimY; sy+=blockDim.y){
for (unsigned int sx = 0; sx < inputsDimX; sx+=blockDim.x) {
const unsigned int inputsIdx =
channel*inputsDimX*inputsDimY + sy*inputsDimX + sx;

unsigned int batchSum = 0;
for(unsigned int batch=0; batch<batchSize; ++batch) {
const unsigned int batchInputOffset = batch * inputSize;
batchSum += firingRate[inputsIdx + batchInputOffset];
}
batchFiringRate[inputsIdx] = batchSum;
}
}
}

}