#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDSoftplus_propagate_kernel(double* x, double* y, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = log(1.0 + exp(x[i]));
}
}