#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDRectifier_backPropagate_kernel(double* x, double* dx, unsigned int size, double leakSlope, int shifting, double clipping)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (shifting > 0)
dx[i] /= (1 << shifting);
else if (shifting < 0)
dx[i] *= (1 << (-shifting));

if (clipping > 0.0) {
dx[i] *= (x[i] > clipping) ? 0.0 : (x[i] > 0.0)
? 1.0
: leakSlope;
}
else
dx[i] *= (x[i] > 0.0) ? 1.0 : leakSlope;
}
}