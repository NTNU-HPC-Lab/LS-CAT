#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateActivity_kernel(int * inputs, char * activity, unsigned int * firingRate, unsigned int * exampleFiringRate, int * totalOutput, unsigned long long int * firstEventTime, unsigned long long int * lastEventTime, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ, unsigned int long long timestamp)
{
const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;

// One batch per block z dimension
const unsigned int batchInputOffset = blockIdx.z * inputSize;

for (unsigned int channel = blockIdx.x; channel < inputsDimZ; channel += gridDim.x) {
for (unsigned int y = threadIdx.y; y < inputsDimY; y += blockDim.y) {
for (unsigned int x = threadIdx.x; x < inputsDimX; x += blockDim.x) {

const unsigned int inputsIdx =
x + y*inputsDimX + channel*inputsDimX*inputsDimY;
int act = inputs[inputsIdx + batchInputOffset];
unsigned int actAbs = abs(act);
char spike = act == 0 ? 0 : act/abs(act);

activity[inputsIdx + batchInputOffset] = spike;
firingRate[inputsIdx + batchInputOffset] += actAbs;
exampleFiringRate[inputsIdx + batchInputOffset] += actAbs;
totalOutput[inputsIdx + batchInputOffset] += act;
}
}
}
}