#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDSaturation_backPropagate_kernel(double* x, double* dx, unsigned int size, int shifting, double threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (shifting > 0)
dx[i] /= (1 << shifting);
else if (shifting < 0)
dx[i] *= (1 << (-shifting));

if (threshold != 0.0) {
dx[i] *= (x[i] > -threshold && x[i] < threshold)
? 1.0 : 0.0;
}
}
}