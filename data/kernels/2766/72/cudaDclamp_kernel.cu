#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDclamp_kernel(double* x, unsigned int size, double minVal, double maxVal)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
x[i] = (x[i] < minVal) ? minVal :
(x[i] > maxVal) ? maxVal :
x[i];
}
}