#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateMostActive_kernel(unsigned int * exampleFiringRate, unsigned int * mostActiveId, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{

const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;

const unsigned int batchInputOffset = blockIdx.z * inputSize;

extern __shared__ unsigned int partialActiveIdx[];

// For case that threadIdx.x > inputSize
partialActiveIdx[threadIdx.x] = 0;

// TODO: Index 0 has a slight advantage here
for (unsigned int i=threadIdx.x; i<inputSize; i+=blockDim.x) {
partialActiveIdx[threadIdx.x] = threadIdx.x;
}

// Search for max ID in each thread
for (unsigned int i=threadIdx.x; i<inputSize; i+=blockDim.x) {
if (exampleFiringRate[i + batchInputOffset] >
exampleFiringRate[partialActiveIdx[threadIdx.x] + batchInputOffset]) {
partialActiveIdx[threadIdx.x] = i;
}
}

__syncthreads();

// Reduction over neurons
for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
if (threadIdx.x < offset){
if (exampleFiringRate[partialActiveIdx[threadIdx.x] + batchInputOffset] <
exampleFiringRate[partialActiveIdx[threadIdx.x + offset] + batchInputOffset]) {
partialActiveIdx[threadIdx.x] =
partialActiveIdx[threadIdx.x + offset];
}
}

__syncthreads();
}

if (threadIdx.x == 0) {
mostActiveId[blockIdx.z] = partialActiveIdx[0];
}

}