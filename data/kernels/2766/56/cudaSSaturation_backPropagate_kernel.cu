#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSaturation_backPropagate_kernel(float* x, float* dx, unsigned int size, int shifting, float threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (shifting > 0)
dx[i] /= (1 << shifting);
else if (shifting < 0)
dx[i] *= (1 << (-shifting));

if (threshold != 0.0f) {
dx[i] *= (x[i] > -threshold && x[i] < threshold)
? 1.0f : 0.0f;
}
}
}