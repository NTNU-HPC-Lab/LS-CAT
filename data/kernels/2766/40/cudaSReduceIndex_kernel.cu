#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSReduceIndex_kernel(  const unsigned int inputSize, const unsigned int inputBatchOffset, const unsigned int outputBatchOffset, const float* valueThreshold, const float* inputs, int* outputMap, float* scores)
{
const int batchPos = blockIdx.z;
const int clsPos = blockIdx.y;

const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

const int inputIndex = index
+ inputSize*clsPos
+ batchPos*inputBatchOffset;

const int outputIndex = index
+ inputSize*clsPos
+ batchPos*outputBatchOffset;

if(index < inputSize)
{

float value = inputs[inputIndex];

if(value >= valueThreshold[clsPos])
{
outputMap[outputIndex] = index;
scores[outputIndex] = value;
}
else
{
outputMap[outputIndex] = -1;
scores[outputIndex] = -1.0;
}

}
}