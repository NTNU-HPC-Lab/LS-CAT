#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSclamp_kernel(float* x, unsigned int size, float minVal, float maxVal)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
x[i] = (x[i] < minVal) ? minVal :
(x[i] > maxVal) ? maxVal :
x[i];
}
}