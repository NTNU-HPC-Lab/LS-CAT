#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDaccumulate_kernel(double* x, int* y, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
x[i] += y[i];
}
}