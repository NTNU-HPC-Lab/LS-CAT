#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSinv_kernel(unsigned int size, const float *x, float *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = 1.0f / x[i];
}
}