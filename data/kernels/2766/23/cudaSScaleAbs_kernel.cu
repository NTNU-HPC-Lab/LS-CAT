#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSScaleAbs_kernel(unsigned int size, float* input, const float scale, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride)
result[i] = fabs(input[i]) * scale + beta * result[i];
}
else {
for (unsigned int i = index; i < size; i += stride)
result[i] = fabs(input[i]) * scale;
}
}