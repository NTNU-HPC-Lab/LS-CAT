#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSAnchorBackPropagateSSD_NegSamples_kernel(const float* inputCls, float* diffOutputsCls, const float* confSamples, const int* keySamples, const int nbSamples, const int nbPositive, const unsigned int nbAnchors, const unsigned int outputsHeight, const unsigned int outputsWidth, const unsigned int batchSize)
{

const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

if(index < nbSamples)
{
const int indexSamples = keySamples[index];
const float error = inputCls[indexSamples];
//diffOutputsCls[indexSamples] = -inputCls[index] / (nbPositive * batchSize);
//printf("error[%d]: %f\n", indexSamples, error);

diffOutputsCls[indexSamples] = -error / (nbPositive * batchSize);
}

}