#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSaccumulate_kernel(float* x, int* y, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
x[i] += y[i];
}
}