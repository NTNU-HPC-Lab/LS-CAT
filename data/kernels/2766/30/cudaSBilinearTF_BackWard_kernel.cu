#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSBilinearTF_BackWard_kernel( unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const float scaleX, const float scaleY, const float* diffInput, float* diffOutputs)
{

const unsigned int inputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*inputWidth*inputHeight;

const unsigned int outputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*outputWidth*outputHeight;
for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x)
{
for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y)
{
const float in_y = oy * scaleY;
const int top_y_index = (int)(floorf(in_y));
//const int bottom_y_index = min((int)(ceilf(in_y)), (int) (inputHeight - 1) ) ;

const int bottom_y_index = (in_y < inputHeight - 1) ? ceilf(in_y) : inputHeight - 1;

const float y_lerp = in_y - top_y_index;
const float inverse_y_lerp = (1.0f - y_lerp);


for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x)
{
const float in_x = ox * scaleX;
const int left_x_index = (int)(floorf(in_x));
//const int right_x_index = min((int)(ceilf(in_x)), (int)(inputWidth - 1));
const int right_x_index = (in_x < inputWidth - 1) ? ceilf(in_x) : inputWidth - 1;


const float x_lerp = in_x - left_x_index;
const float inverse_x_lerp = (1.0f - x_lerp);

const unsigned int inLeftTopIdx = left_x_index + top_y_index*inputWidth + ch*inputWidth*inputHeight + inputOffset;
const unsigned int inRightTopIdx = right_x_index + top_y_index*inputWidth + ch*inputWidth*inputHeight + inputOffset;
const unsigned int inLeftBotIdx = left_x_index + bottom_y_index*inputWidth + ch*inputWidth*inputHeight + inputOffset;
const unsigned int inRightBotIdx = right_x_index + bottom_y_index*inputWidth + ch*inputWidth*inputHeight + inputOffset;

const unsigned int outIdx = ox + oy*outputWidth + ch*outputWidth*outputHeight + outputOffset;
const float outData = diffInput[outIdx];

diffOutputs[inLeftTopIdx]  += outData * inverse_y_lerp * inverse_x_lerp ;
diffOutputs[inRightTopIdx]  += outData * inverse_y_lerp * x_lerp ;
diffOutputs[inLeftBotIdx]  += outData * y_lerp * inverse_x_lerp ;
diffOutputs[inRightBotIdx]  += outData * y_lerp * x_lerp ;

}
}
}
}