#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateMostActive_kernel(unsigned int * exampleIds, unsigned int * exampleFiringRate, unsigned int * mostActiveId, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{

const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;

const unsigned int batchInputOffset = blockIdx.z * inputSize;

const unsigned int blockOffset = blockIdx.x * blockDim.x;

const unsigned int partialIdx = threadIdx.x + blockOffset;

// TODO: Also used shared memory for firing rates to avoid global
// memory accesses
extern __shared__ unsigned int partialActiveIdx[];

// TODO: Index 0 has a slight advantage here
partialActiveIdx[threadIdx.x] = 0;
if (partialIdx < inputSize){
partialActiveIdx[threadIdx.x] = exampleIds[partialIdx + batchInputOffset];
}

__syncthreads();

// Reduction over neurons
for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
if (threadIdx.x < offset){
if (exampleFiringRate[partialActiveIdx[threadIdx.x]] <
exampleFiringRate[partialActiveIdx[threadIdx.x + offset]]) {
partialActiveIdx[threadIdx.x] =
partialActiveIdx[threadIdx.x + offset];
}
}

__syncthreads();
}

if (threadIdx.x == 0) {
mostActiveId[blockIdx.x+gridDim.x*blockIdx.z] = partialActiveIdx[0];
}

}