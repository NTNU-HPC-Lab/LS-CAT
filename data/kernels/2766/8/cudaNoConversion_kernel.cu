#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaNoConversion_kernel(float * data, float * tickOutputsTraces, float * tickOutputsTracesLearning, float scaling, unsigned int inputDimX, unsigned int inputDimY, unsigned int inputDimZ)
{
const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
const unsigned int batchOffset = blockIdx.x * inputSize;

for (unsigned int idx = threadIdx.x; idx < inputSize; idx += blockDim.x) {
float value = data[idx + batchOffset];
tickOutputsTraces[idx + batchOffset] = scaling*value;
tickOutputsTracesLearning[idx + batchOffset] += scaling*value;
}
}