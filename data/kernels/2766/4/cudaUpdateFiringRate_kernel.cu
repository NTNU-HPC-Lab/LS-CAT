#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateFiringRate_kernel(int * firingRate, int * totalFiringRatePartial, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{

const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;

const unsigned int batchInputOffset = blockIdx.z * inputSize;

const unsigned int blockOffset = blockIdx.x * blockDim.x;

const unsigned int partialIdx = threadIdx.x + blockOffset;

extern __shared__ unsigned int partialSum[];

// Perform first level of reduction during initialization
// This is more efficient since we need all threads to load data
// but the partial sum will see only half of the threads active
//partialSum[threadIdx.x] = firingRate[partialIdx + batchInputOffset] +
//    firingRate[partialIdx + blockDim.x + batchInputOffset];

partialSum[threadIdx.x] = 0;
if (partialIdx < inputSize){
partialSum[threadIdx.x] = firingRate[partialIdx + batchInputOffset];
}

__syncthreads();

// Reduction over neurons
for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
if (threadIdx.x < offset){
partialSum[threadIdx.x] += partialSum[threadIdx.x + offset];
}

__syncthreads();
}

if (threadIdx.x == 0) {
totalFiringRatePartial[blockIdx.x+gridDim.x*blockIdx.z] = partialSum[0];
}


}