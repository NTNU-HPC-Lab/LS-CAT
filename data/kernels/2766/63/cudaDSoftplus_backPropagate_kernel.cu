#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDSoftplus_backPropagate_kernel(double* x, double* dx, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
dx[i] *= (1.0 - exp(-x[i]));
}
}