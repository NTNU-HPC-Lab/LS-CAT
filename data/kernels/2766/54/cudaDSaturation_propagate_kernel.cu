#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDSaturation_propagate_kernel(double* x, double* y, unsigned int size, int shifting, double threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
double value = x[i];

if (shifting > 0)
value /= (1 << shifting);
else if (shifting < 0)
value *= (1 << (-shifting));

if (threshold != 0.0) {
y[i] = (value < -threshold) ? -threshold
: (value > threshold) ? threshold
: value;
}
}
}