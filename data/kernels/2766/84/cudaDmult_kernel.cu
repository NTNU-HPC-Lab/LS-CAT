#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDmult_kernel(unsigned int size, const double *x1, const double *x2, double *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = x1[i] * x2[i];
}
}