#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDRectifier_propagate_kernel(double* x, double* y, unsigned int size, double leakSlope, int shifting, double clipping)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
double value = x[i];

if (shifting > 0)
value /= (1 << shifting);
else if (shifting < 0)
value *= (1 << (-shifting));

if (clipping > 0.0)
y[i] = (value > 0.0) ? min(value, clipping) : leakSlope * value;
else
y[i] = (value > 0.0) ? value : leakSlope * value;
}
}