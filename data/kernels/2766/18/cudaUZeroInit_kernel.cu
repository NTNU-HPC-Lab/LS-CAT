#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUZeroInit_kernel(unsigned int size, unsigned int* data)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride)
data[i] = 0U;
}