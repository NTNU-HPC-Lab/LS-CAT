#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSEuclideanSumBackward_kernel(unsigned int size, float* diffInput, float* input, float* output, const float scale, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride) {
result[i] = (output[i] != 0.0f)
? diffInput[i] * scale * (input[i] / output[i]) + beta * result[i]
: beta * result[i];
}
}
else {
for (unsigned int i = index; i < size; i += stride) {
result[i] = (output[i] != 0.0f)
? diffInput[i] * scale * (input[i] / output[i])
: 0.0f;
}
}
}