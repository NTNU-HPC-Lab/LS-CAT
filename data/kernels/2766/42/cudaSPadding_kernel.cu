#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSPadding_kernel( unsigned int nbOutputs, unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, int leftPad, int rightPad, int topPad, int botPad, const float* input, float* outputs)
{

const unsigned int inputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*inputWidth*inputHeight;

const unsigned int outputOffset
= (blockIdx.z * blockDim.z + threadIdx.z) * nbOutputs*outputWidth*outputHeight;

// nbCh = nbChannels for propagate
//      = nbOutputs for back-propagate
const unsigned int nbCh = min(nbChannels, nbOutputs);

for (unsigned int ch = blockIdx.x; ch < nbCh; ch += gridDim.x)
{
for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y)
{
for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x)
{
float outputValue = 0.0;
int ix = (int) ox - leftPad;
int iy = (int) oy - topPad;

if( ix >= 0 && ix < (int) inputWidth
&& iy >= 0 && iy < (int) inputHeight )
{
outputValue = input[ix +
iy*inputWidth
+ ch*inputWidth*inputHeight
+ inputOffset];

}
outputs[ ox + oy*outputWidth
+ ch*outputWidth*outputHeight + outputOffset]  = outputValue;

}
}
}


}