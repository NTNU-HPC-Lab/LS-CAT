#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDquantize_kernel(double* x, double* y, unsigned int size, double minVal, double maxVal, unsigned int quantizationLevels, bool truncate)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (quantizationLevels > 1) {
const double scaling = (maxVal - minVal)
/ (double)(quantizationLevels - 1);

for (unsigned int i = index; i < size; i += stride) {
const double clamped = (x[i] < minVal) ? minVal :
(x[i] > maxVal) ? maxVal :
x[i];

if (truncate)
y[i] = (int)((clamped - minVal) / scaling) * scaling + minVal;
else {
y[i] = (int)round((clamped - minVal) / scaling)
* scaling + minVal;
}
}
}
else {
for (unsigned int i = index; i < size; i += stride)
y[i] = ((x[i] >= 0.0) ? 1.0 : -1.0);
}
}