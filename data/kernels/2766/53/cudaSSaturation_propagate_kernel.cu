#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSaturation_propagate_kernel(float* x, float* y, unsigned int size, int shifting, float threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
float value = x[i];

if (shifting > 0)
value /= (1 << shifting);
else if (shifting < 0)
value *= (1 << (-shifting));

if (threshold != 0.0f) {
y[i] = (value < -threshold) ? -threshold
: (value > threshold) ? threshold
: value;
}
}
}