#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul(int* A, int* B, int* C){
int col = blockIdx.x * blockDim.x + threadIdx.x;
int lig = blockIdx.y * blockDim.y + threadIdx.y;

int index = lig * N + col;

if (col < N && lig < N){
int inter = 0;
for (int i = 0; i<N; ++i){
inter += A[lig*N + i] * B[i*N + col];
}
C[index] = inter;
}
}