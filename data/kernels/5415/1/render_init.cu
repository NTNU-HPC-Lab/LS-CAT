#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if ((i >= max_x) || (j >= max_y)) return;
int pixel_index = j*max_x + i;
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}