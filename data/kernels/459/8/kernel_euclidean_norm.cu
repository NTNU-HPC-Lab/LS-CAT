#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_euclidean_norm(const double      *vec, int                 numElements, double            *answer)
{
extern __shared__ double square[]; // one element per thread

int i = threadIdx.x; // numElements assumed to fit into one block
square[i] = vec[i] * vec[i];

__syncthreads();

if (i == 0) {
double sum = 0;
for (int j = 0; j < numElements; ++j) {
sum += square[j];
}
*answer = sqrt(sum);
}
}