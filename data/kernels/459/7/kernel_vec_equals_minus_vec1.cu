#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_vec_equals_minus_vec1(double      *vec, double      *vec1, int numElements)
{

int iam = threadIdx.x;
int bid = blockIdx.x;
int threads_in_block = blockDim.x;
int gid = bid*threads_in_block + iam;

if (gid < numElements){
vec[gid] = -vec1[gid];
}
}