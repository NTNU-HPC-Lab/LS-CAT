#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_dot_product(const double * vec1, const double * vec2, int             numElements, double       * answer)
{
extern __shared__ double products[]; // one element per thread

int i = threadIdx.x; // numElements assumed to fit into one block
products[i] = vec1[i] * vec2[i];

__syncthreads();

if (i == 0) {
double sum = 0;
for (int j = 0; j < numElements; ++j) {
sum += products[j];
}
*answer = sum;
}
}