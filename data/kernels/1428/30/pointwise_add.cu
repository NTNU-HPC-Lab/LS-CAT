#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pointwise_add(float *d_res, const float *d_op1, const float *d_op2, const int len)
{
const int pos = blockIdx.x*blockDim.x + threadIdx.x;

if(pos >= len) return;

d_res[pos] = d_op1[pos] + d_op2[pos];
}