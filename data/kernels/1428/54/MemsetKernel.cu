#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
int i = threadIdx.y + blockDim.y * blockIdx.y;
int j = threadIdx.x + blockDim.x * blockIdx.x;

if (i >= h || j >= w) return;

const int pos = i * w + j;

image[pos] = value;
}