#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DeviceMultiply(double* left, double* right, double* result, int left_rows, int left_cols, int right_cols) {
int i = threadIdx.y;
int j = threadIdx.x;
int x_stride = blockDim.x;
int y_stride = blockDim.y;
__shared__ double temp[16][16];
for (int y_offset = 0; i + y_offset < left_rows; y_offset += y_stride) {
for (int x_offset = 0; j + x_offset < right_cols; x_offset += x_stride) {
temp[i][j] = 0.0;
for (int k = 0; k < left_cols; ++k) {
int left_idx = (y_offset + i) * left_cols + k;
int right_idx = k * right_cols + x_offset + j;
temp[i][j] += left[left_idx] * right[right_idx];
}
int result_idx = (y_offset + i) * right_cols + x_offset + j;
result[result_idx] = temp[i][j];
}
}
}