#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int sign(DECNUM x)
{
return((x > 0.0f) - (x < 0.0f));
}
__device__ int mminus2(int ix, int nx)
{
int xminus;
if (ix <= 1)
{
xminus = 0;
}
else
{
xminus = ix - 2;
}
return(xminus);
}
__device__ int pplus(int ix, int nx)
{
int xplus;
if (ix == nx - 1)
{
xplus = nx - 1;
}
else
{
xplus = ix + 1;
}
return(xplus);

}
__device__ int mminus(int ix, int nx)
{
int xminus;
if (ix == 0)
{
xminus = 0;
}
else
{
xminus = ix - 1;
}
return(xminus);
}
__global__ void latbnd(int nx, int ny, DECNUM * uu)
{
unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
unsigned int i = ix + iy*nx;
int tx = threadIdx.x;
int ty = threadIdx.y;

__shared__ DECNUM uut[16][16];
__shared__ DECNUM uub[16][16];

if (ix < nx && iy < ny)
{
unsigned int yminus = mminus(iy, ny);
unsigned int yminus2 = mminus2(iy, ny);
unsigned int yplus = pplus(iy, ny);


uut[tx][ty] = uu[ix + yplus*nx];
uub[tx][ty] = uu[ix + yminus*nx];

if (iy == 0)
{
uu[i] = uut[tx][ty];
}
if (iy == ny - 1)
{
uu[i] = uub[tx][ty];
}
}
//
}