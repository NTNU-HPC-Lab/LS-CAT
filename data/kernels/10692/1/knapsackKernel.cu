#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k){

int c = blockIdx.x*512 + threadIdx.x;
if(c<c_min || c>capacity){return;}
if(input_f[c] < input_f[c-weights[k-1]]+profits[k-1]){
output_f[c] = input_f[c-weights[k-1]]+profits[k-1];
}
else{
output_f[c] = input_f[c];
}
}