#include "hip/hip_runtime.h"
#include "includes.h"

#define index(i, j, N)  ((i)*(N+1)) + (j)

__device__ int maximum(int a, int b) {
return (a > b)? a : b;
}
__global__ void knapsackKernel(int *profits, int *weights, int *f, int capacity, int i){

int c = threadIdx.x;

if(i==0 || c==0)	f[index(i,c,capacity)] = 0;
else if(weights[i-1] <= c){
f[index(i,c,capacity)] = maximum(f[index(i-1,c,capacity)], profits[i-1]+f[index(i-1,c-weights[i-1],capacity)]);
}
else
f[index(i,c,capacity)] = f[index(i-1,c,capacity)];
}