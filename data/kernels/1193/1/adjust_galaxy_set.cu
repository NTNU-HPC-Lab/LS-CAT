#include "hip/hip_runtime.h"
#include "includes.h"

#define BIN_WIDTH 0.25
#define BLOCK_DIM 256
#define COVERAGE 180
#define LINE_LENGTH 30

#define BINS_TOTAL (COVERAGE * (int)(1 / BIN_WIDTH))

typedef struct Galaxy
{
float declination;
float declination_cos;
float declination_sin;
float right_ascension;
} Galaxy;


__device__ float arcminutes_to_radians(float arcminute_value)
{
return (M_PI * arcminute_value) / (60 * 180);
}
__global__ void adjust_galaxy_set(Galaxy *galaxy_set, int n)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int i = index; i < n; i += stride)
{
float declination = arcminutes_to_radians(galaxy_set[i].declination);
galaxy_set[i].declination = declination;
galaxy_set[i].declination_cos = cosf(declination);
galaxy_set[i].declination_sin = sinf(declination);

galaxy_set[i].right_ascension = arcminutes_to_radians(galaxy_set[i].right_ascension);
}
}