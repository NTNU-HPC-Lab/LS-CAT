#include "hip/hip_runtime.h"
#include "includes.h"

#define BIN_WIDTH 0.25
#define BLOCK_DIM 256
#define COVERAGE 180
#define LINE_LENGTH 30

#define BINS_TOTAL (COVERAGE * (int)(1 / BIN_WIDTH))

typedef struct Galaxy
{
float declination;
float declination_cos;
float declination_sin;
float right_ascension;
} Galaxy;


__global__ void measure_galaxy_distribution(int *DD_histogram, int *DR_histogram, int *RR_histogram, float *distribution, int n)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int i = index; i < n; i += stride)
{
if (RR_histogram[i] == 0)
continue;

distribution[i] = (DD_histogram[i] - 2.0f * DR_histogram[i] + RR_histogram[i]) / RR_histogram[i];
}
}