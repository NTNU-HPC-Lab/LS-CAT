#include "hip/hip_runtime.h"
#include "includes.h"



__global__ static void sum_channels(float *dest, const float *src, uint channels, uint num_channel_elem)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= num_channel_elem)
return;

float acc = 0;
for (uint i = 0; i < channels; ++i)
acc += src[idx + i * num_channel_elem];
dest[idx] = acc;
}