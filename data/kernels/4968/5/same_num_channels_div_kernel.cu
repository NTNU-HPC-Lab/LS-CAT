#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void same_num_channels_div_kernel(const float *data_l, const float *data_r, float *result, unsigned total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

if (idx / 2 < total) {
result[idx] = (data_l[idx] * data_r[idx] + data_l[idx + 1] * data_r[idx + 1]) /
(data_r[idx] * data_r[idx] + data_r[idx + 1] * data_r[idx + 1]);
result[idx + 1] = (data_l[idx + 1] * data_r[idx] - data_l[idx] * data_r[idx + 1]) /
(data_r[idx] * data_r[idx] + data_r[idx + 1] * data_r[idx + 1]);
}
}