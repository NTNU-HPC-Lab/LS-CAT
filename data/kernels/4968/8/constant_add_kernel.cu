#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void constant_add_kernel(const float *data_l, float constant, float *result, int total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

if (idx / 2 < total) {
result[idx] = data_l[idx] + constant;
result[idx + 1] = data_l[idx + 1];
}
}