#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectFill( int * data1, int * data2, int * restult, unsigned long sizeOfArray )
{
unsigned long i = blockDim.x * blockIdx.x + threadIdx.x;
if( i < sizeOfArray )
{
restult[ i ] = data1[i] + data2[i];
}
}