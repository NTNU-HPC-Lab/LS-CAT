#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;
__device__ __managed__ float weightedinputs[25]; // used as list of input neuron
__device__ __managed__ float weights[25]; // used as list of neuron conection weigths
__device__ __managed__ int inputs[25]; // used as list of neuron conection weigths
__device__ __managed__ float output = 0; // used to return output
__device__ __managed__ int expctd = 0; // used to return output



__global__ void feedbackward(){			// trains the weights
float lr = 0.3;
float error = (expctd - output);
weights[threadIdx.x] = weights[threadIdx.x] + error * inputs[threadIdx.x] * lr;
}