#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGlobalCol(float *out, float *in, const int nx, const int ny)
{
unsigned int i = threadIdx.x+blockDim.x*blockIdx.x;
unsigned int j = threadIdx.y+blockDim.y*blockIdx.y;

if (i<nx && j<ny)
{
out[i*ny+j] = in[i*ny+j];
}
}