#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGlobalRow(float *in, float *out, const int nx, const int ny)
{
unsigned int i = threadIdx.x+blockDim.x*blockIdx.x;
unsigned int j = threadIdx.y+blockDim.y*blockIdx.y;

if (i<nx && j<ny)
{
out[j*nx+i] = in[j*nx+i];
}
}