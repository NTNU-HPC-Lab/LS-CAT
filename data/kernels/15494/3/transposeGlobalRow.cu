#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeGlobalRow(float *in, float *out, const int nx, const int ny)
{
unsigned int i = threadIdx.x+blockDim.x*blockIdx.x;
unsigned int j = threadIdx.y+blockDim.y*blockIdx.y;

if (i<nx && j<ny)
{
out[i*ny+j] = in[j*nx+i];
}
}