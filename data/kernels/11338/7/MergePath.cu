#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int satisfies(int i, int j, int *A, int *B)
{
return (A[i] <= B[j]);
}
__global__ void MergePath(int *A, int *B, int* C, int *x, int *y, int n)
{
int num_of_threads = blockDim.x;
int idx = threadIdx.x;
bool flag = false;
if (idx == 0)
{
x[idx] = 0;
y[idx] = 0;
flag = true;
}
int A_start = idx*(2 * n) / num_of_threads; //only when len(A)==len(B)
int B_start = max(0, A_start - (n - 1));
A_start = min(n - 1, A_start);
int length_of_array;

if (B_start == 0)
{

length_of_array = A_start + 1;
}
else
length_of_array = n - B_start;

int left = 0, right = length_of_array - 1;
// cout<<A_start<<" "<<B_start<<" "<<length_of_array<<endl<<"-------------------------------------------\n";

while (left <= right && !flag)
{
// cout<<left<<" "<<right<<endl;
int mid = left + (right - left) / 2;
int I = A_start - mid;
int J = B_start + mid;
if (!satisfies(I, J, A, B))
{
left = mid + 1;
}
else
{
if (J == 0)
{
x[idx] = (I + 1);
y[idx] = (J);
flag = true;
}
else if (I == n - 1)
{
x[idx] = (I + 1);
y[idx] = (J);
flag = true;
}
else
{
if (!satisfies(I + 1, J - 1, A, B))
{
x[idx] = (I + 1);
y[idx] = (J);
flag = true;
}
else
{
right = mid;
}
}
}
}
left--;
if (!flag)
{
x[idx] = (A_start - left);
y[idx] = (n);
}
__syncthreads();

int end_x, end_y;
if (idx == num_of_threads - 1)
{
end_x = n;
end_y = n;
}
else
{
end_x = x[idx + 1];
end_y = y[idx + 1];
}
int cur_x = x[idx];
int cur_y = y[idx];
int put_at = cur_x + cur_y;
while (cur_x<end_x && cur_y<end_y)
{
if (A[cur_x] <= B[cur_y])
{
C[put_at++] = A[cur_x++];
}
else
{
C[put_at++] = B[cur_y++];
}
}
while (cur_x<end_x)
C[put_at++] = A[cur_x++];
while (cur_y<end_y)
C[put_at++] = B[cur_y++];
}