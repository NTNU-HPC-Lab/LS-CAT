#include "hip/hip_runtime.h"
#include "includes.h"
#define PI 3.141592653589793
#define BLOCKSIZE 1024


__global__ void cuMultiplyNum(float *dst, float num, int size)
{
int id=blockIdx.x*blockDim.x+threadIdx.x;
if(id>=size) return;
dst[id]*=num;
}