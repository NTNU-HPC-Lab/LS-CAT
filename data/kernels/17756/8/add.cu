#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float* vec_a, float* vec_b, float* vec_c, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n)
{
vec_c[i] = vec_a[i] + vec_b[i];
i += blockDim.x * gridDim.x;
}
}