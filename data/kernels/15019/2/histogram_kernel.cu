#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_kernel(float* magnitude, float* phase, float* histograms, int input_width, int input_height, int cell_grid_width, int cell_grid_height, int magnitude_step, int phase_step, int histograms_step, int cell_width, int cell_height, int num_bins)
{
//TODO: make the buffer sizes dependent on an input or template parameter.
// Each thread block needs to store intermediate results for 64 gradients
// and also 8 different histograms, each with 9 bins.
__shared__ int s_lbin_pos[64];
__shared__ float s_lbin[64];
__shared__ int s_rbin_pos[64];
__shared__ float s_rbin[64];
__shared__ float s_hist[9 * 8];

// The columns of the image are mapped to the first dimension of the block
// grid and the first dimension of the thread block.
int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
// If current position is outside the image, stop here
if(pixel_x >= input_width)
{
return;
}
// The columns of the image are mapped to the second dimension of the block
// grid and the second dimension of the thread block.
int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
// If current position is outside the image, stop here
if(pixel_y >= input_height)
{
return;
}

// Each row has magnitude_step size
int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
// Each row has phase_step size
int phase_pixel_idx = pixel_y * phase_step + pixel_x;

// The phase was previously normalized to [0,1]
float bin_size = 1.0f / (float)num_bins;
// By dividing by the bin size and taking the integer part, you find out
// inside which bin the gradient is at. If it's greater than the middle of the bin
// it will be divided between this one and the next, if it's lesser it will
// be divided between this and the previous one. By subtracting 0.5 before
// taking the integer part, the division will always be between this bin and
// the next.
int left_bin = (int)floor((phase[phase_pixel_idx] / bin_size) - 0.5f);
// The result of the previous operation might be negative. If so, the next
// bit fixes that. Otherwise that changes nothing.
left_bin = (left_bin + num_bins) % num_bins;
// Take the next bin as the right bin.
// If the left bin is the last one, this will be outside range. Wait a bit
// before taking the remainder, because this value needs to be used in the
// formula below.
int right_bin = (left_bin + 1);
// Calculate the distance between the gradient phase and the limit between
// the left and right bins. Normalized by the bin size, the limit is equal
// to the right bin identifier.
float delta = (phase[phase_pixel_idx] / bin_size) - right_bin;
if(delta < -0.5)
{
delta += num_bins;
}
//Fix range for right_bin now
right_bin = right_bin % num_bins;

// Store the bin positions and amounts for each bin on shared buffers.
s_lbin_pos[threadIdx.x] = left_bin;
s_lbin[threadIdx.x] = (0.5 - delta) * magnitude[mag_pixel_idx];
s_rbin_pos[threadIdx.x] = right_bin;
s_rbin[threadIdx.x] = (0.5 + delta) * magnitude[mag_pixel_idx];

// Wait for other threads.
__syncthreads();

// Initialize histograms shared buffer.
s_hist[threadIdx.x] = 0.0f;
if(threadIdx.x < 8)
{
s_hist[threadIdx.x + 64] = 0.0f;
}

int cell_y = pixel_y / cell_height;

// Each partial histogram will be calculated by only one thread.
if(threadIdx.x < 8)
{
int s_hist_idx = 9 * threadIdx.x;
for(int i = 1; i < 8; ++i)
{
s_hist[s_hist_idx + s_lbin_pos[8 * threadIdx.x + i]] += s_lbin[8
* threadIdx.x + i];
s_hist[s_hist_idx + s_rbin_pos[8 * threadIdx.x + i]] += s_rbin[8
* threadIdx.x + i];
}
}

// Wait until all threads finish.
__syncthreads();

// Add to the complete histogram sum using atomic operations.
int out_idx = cell_y * histograms_step + threadIdx.x;
atomicAdd(&(histograms[out_idx]), s_hist[threadIdx.x]);

if(threadIdx.x < 8)
{
atomicAdd(&(histograms[out_idx + 64]), s_hist[threadIdx.x + 64]);
}
}