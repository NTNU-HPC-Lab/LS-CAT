#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void block_normalization_kernel(float* histograms, float* descriptor, int histograms_step, int block_grid_width, int block_grid_height, int block_width, int block_height, int num_bins, int cell_grid_width, int block_stride_x, int block_stride_y)
{
//TODO: make the buffer sizes dependent on an input or template parameter.
// Each thread block will process 8 hog blocks. Each hog block has 4 cells.
// Each cell has 9 bins.
__shared__ float s_blocks[9 * 4 * 8];
__shared__ float L1_norm[8];
int block_x = blockIdx.x * 8 + threadIdx.z;
if(block_x >= block_grid_width)
{
return;
}
int block_y = blockIdx.y;
if(block_y >= block_grid_height)
{
return;
}
int block_idx = block_y * blockDim.y + block_x;
int cell_x = block_x * block_stride_x + threadIdx.y % 2;
int cell_y = block_y * block_stride_y + threadIdx.y / 2;
int hist_idx = histograms_step * cell_y + num_bins * (cell_x) + threadIdx.x;

int s_blocks_idx = 9 * threadIdx.y + threadIdx.x;
s_blocks[s_blocks_idx] = histograms[hist_idx];

__syncthreads();

int thread_id = 36 * threadIdx.z + 9 * threadIdx.y + threadIdx.x;
int elements_per_block = block_height * block_width * num_bins;
if(thread_id < 8)
{
L1_norm[thread_id] = 0.0f;
for(int i = 0; i < elements_per_block; ++i)
{
L1_norm[thread_id] += s_blocks[elements_per_block * thread_id + i];
}
}

__syncthreads();

descriptor[elements_per_block * block_idx + s_blocks_idx] =
s_blocks[s_blocks_idx] / L1_norm[threadIdx.z];
}