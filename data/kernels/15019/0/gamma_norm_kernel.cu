#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gamma_norm_kernel(float* img, int image_height, int image_width, int image_step)
{
// The thread block has size (3,n). The first dimension of the thread block
// corresponds to color channels.
int channel = threadIdx.x;
// The columns of the image are mapped to the first dimension of the block
// grid, but to the second dimension of the thread block, as the first
// already corresponds to color channels.
int pixel_x = blockIdx.x * blockDim.y + threadIdx.y;
// If current position is outside the image, stop here
if(pixel_x >= image_width)
{
return;
}
// The columns of the image are mapped to the second dimension of the block
// grid, but to the third dimension of the thread block.
int pixel_y = blockIdx.y * blockDim.z + threadIdx.z;
// If current position is outside the image, stop here
if(pixel_y >= image_height)
{
return;
}

// Each row has image_step pixels and each pixel has three channels
int in_pixel_idx = pixel_y * image_step + pixel_x * 3 + channel;

// Finally perform the normalization
img[in_pixel_idx] = sqrt(img[in_pixel_idx] / 256.0f);

}