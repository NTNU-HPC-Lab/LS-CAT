#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_preact_fc(float* input, float* preact, float* weight, const int size, const int in_channel, const int out_channel)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int weight_channel = in_channel * out_channel;
const int N = out_channel * in_channel * size * size;  // number of elements of weight matrix

for (int n = N * pos / totalPos; n < N * (pos+1) / totalPos; ++n) {
int idx = n;
const int i_channel = ((idx /= 1	) % weight_channel);
const int i_row = ((idx /= weight_channel	) % size);
const int i_col = ((idx /= size	) % size);

atomicAdd(&preact[i_channel % out_channel], weight[(i_channel * size + i_col) * size + i_row] * input[((i_channel % in_channel) * size + i_col) * size + i_row]);
}
}