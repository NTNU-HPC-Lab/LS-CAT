#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumGrad(float* input1, float* input2, float* input3, float* input4, float* output, const int numElem)
{
size_t pos = blockDim.x * blockIdx.x + threadIdx.x;
size_t size = blockDim.x * gridDim.x;

for(int i = numElem * pos / size; i < numElem * (pos+1) / size; i++){
output[i] = input1[i] + input2[i] + input3[i] + input4[i];
}
}