#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_bias_conv(float* preact, float* bias, const int size, const int n_channel)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int N = n_channel * size * size;

for (int n = N * pos / totalPos; n < N * (pos+1) / totalPos; ++n) {
int idx = n;
const int i_channel = ((idx /= 1	) % n_channel);
const int i_row = ((idx /= n_channel	) % size);
const int i_col = ((idx /= size	) % size);

preact[(i_channel * size + i_col) * size + i_row] += bias[i_channel];
}
}