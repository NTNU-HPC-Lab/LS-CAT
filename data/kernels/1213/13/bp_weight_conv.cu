#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_weight_conv(float* d_weight, float* d_preact, float* p_output, const int kernel_size, const int size, const int n_size, const int in_channel, const int out_channel, bool SAME)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;
const int N = kernel_size * kernel_size * n_size * n_size * in_channel * out_channel;  // total number of connections in this convolution
const int weight_channel = in_channel * out_channel;  // actual number of channels of weight matrix
const int padding = (kernel_size - 1) / 2;  // number of padding for both ends
int input_row, input_col;

// distribute certain number of connections to each thread regardless of detailed position and shape
for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n++){
int idx = n;
const int i_kernel_row = ((idx /= 1	) % kernel_size);
const int i_kernel_col = ((idx /= kernel_size	) % kernel_size);
const int i_channel = ((idx /= kernel_size	) % weight_channel);
const int i_row = ((idx /= weight_channel	) % n_size);
const int i_col = ((idx /= n_size	) % n_size);

// corresponding position of the input matrix
if (SAME){ // SAME padding scheme implemented
input_row = i_kernel_row + i_row - padding;
input_col = i_kernel_col + i_col - padding;
}
else{
input_row = i_kernel_row + i_row;
input_col = i_kernel_col + i_col;
}
if(input_row >= 0 && input_row < size && input_col >=0 && input_col < size){
atomicAdd(&d_weight[(i_channel * kernel_size + i_kernel_col) * kernel_size + i_kernel_row],
d_preact[((i_channel % out_channel) * n_size + i_col) * n_size + i_row] * p_output[((i_channel % in_channel) * size + input_col) + input_row]);
}
}
}