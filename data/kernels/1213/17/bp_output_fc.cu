#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_output_fc(float *d_output, float *d_preact, float *weight, const int size, const int in_channel, const int out_channel)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int N = out_channel * in_channel * size * size;
const int weight_channel = out_channel * in_channel;

for (int n = N * pos / totalPos; n < N * (pos+1) / totalPos; ++n) {
int idx = n;
const int i_channel = ((idx /= 1	) % weight_channel);
const int i_row = ((idx /= weight_channel	) % size);
const int i_col = ((idx /= size	) % size);

atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row], d_preact[i_channel % out_channel] * weight[(i_channel * size + i_col) * size + i_row]);
}
}