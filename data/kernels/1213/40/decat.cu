#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decat(float* output1, float* output2, float* output3, float* output4, size_t num1, size_t num2, size_t num3, size_t num4, size_t maxNum, float* input, const int numPerBatch)
{
size_t i = blockDim.x * blockIdx.x + threadIdx.x;


for(;i < maxNum; i += size_t(blockDim.x * gridDim.x)){
size_t batchIdx = i / numPerBatch; // which batch this thread is working in
const int batchOffset = i - batchIdx * numPerBatch; // offset of current thread in current batch

if(batchOffset < num1){  // first output
output1[batchOffset + batchIdx * num1] = input[i];
}
else if(batchOffset < (num1 + num2)){  // second output
output2[(batchOffset - num1) + batchIdx * num2] = input[i];
}
else if(batchOffset < (num1 + num2 + num3)){  // third input
output3[(batchOffset - (num1 + num2)) + batchIdx * num3] = input[i];
}
else{  // fourth input
output4[(batchOffset - (num1 + num2 + num3)) + batchIdx * num4] = input[i];
}
}
}