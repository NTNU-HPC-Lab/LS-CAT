#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void concat(float* output, float* input1, float* input2, float* input3, float* input4, const int size, const int in_channel1, const int in_channel2, const int in_channel3, const int in_channel4)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int out_channel = in_channel1 + in_channel2 + in_channel3 + in_channel4;  // # of channel for output
const int N = size * size;  // total elements per channel

if(pos < N){
for(int n = 0; n < out_channel; n++){
const int row = pos / size;
const int col = pos % size;
if(n < in_channel1){  // first input
output[(n * size + col) * size + row] = input1[(n * size + col) * size + row];
}
else if(n < in_channel1 + in_channel2){  // second input
output[(n * size + col) * size + row] = input2[((n - in_channel1) * size + col) * size + row];
}
else if(n < in_channel1 + in_channel2 + in_channel3){  // third input
output[(n * size + col) * size + row] = input3[((n - in_channel1 - in_channel2) * size + col) * size + row];
}
else{  // last input
output[(n * size + col) * size + row] = input4[((n - in_channel1 - in_channel2 - in_channel3) * size + col) * size + row];
}
}
}
}