#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_maxpool(float* output, float* input, const int kernel_size, const int size, const int n_size, const int in_channel, bool SAME)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;
const int N = kernel_size * kernel_size * n_size * n_size * in_channel;  // total number of connections in this convolution
const int padding = (kernel_size - 1) / 2;  // number of padding for both ends
int input_row, input_col;
// distribute certain number of connections to each thread regardless of detailed position and shape
for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n++){
int idx = n;
const int i_kernel_row = ((idx /= 1	) % kernel_size);
const int i_kernel_col = ((idx /= kernel_size	) % kernel_size);
const int i_channel = ((idx /= kernel_size	) % in_channel);
const int i_row = ((idx /= in_channel	) % n_size);
const int i_col = ((idx /= n_size	) % n_size);
float maxidx = (float)-100;
// corresponding position of the input matrix and size of output matrix
if (SAME){ // SAME padding scheme implemented
input_row = i_kernel_row + i_row - padding;
input_col = i_kernel_col + i_col - padding;
}
else{
input_row = i_kernel_row + i_row;
input_col = i_kernel_col + i_col;
}
if(input_row >= 0 && input_row < size && input_col >=0 && input_col < size){
if (input[((i_channel % in_channel) * size + input_col) * size + input_row] > maxidx)
output[((i_channel % in_channel) * n_size + i_col) * n_size + i_row] = input[((i_channel % in_channel) * size + input_col) * size + input_row];
}
}
}