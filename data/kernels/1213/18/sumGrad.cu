#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumGrad(float* output, float* input1, float* input2, float* input3, float* input4, const int numElem)
{
size_t pos = blockDim.x * blockIdx.x + threadIdx.x;
size_t size = blockDim.x * gridDim.x;

for(int i = numElem * pos / size; i < numElem * (pos+1) / size; i++){
output[i] = input1[i] + input2[i] + input3[i] + input4[i];
}
}