#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_weight_fc(float *d_weight, float *d_preact, float *p_output, const int size, const int in_channel, const int out_channel)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int N = out_channel * in_channel * size * size;
const int weight_channel = out_channel * in_channel;

for (int n = N * pos / totalPos; n < N * (pos+1) / totalPos; ++n) {
int idx = n;
const int i_channel = ((idx /= 1	) % weight_channel);
const int i_row = ((idx /= weight_channel	) % size);
const int i_col = ((idx /= size	) % size);

d_weight[(i_channel * size + i_col) * size + i_row] = d_preact[i_channel % out_channel] * p_output[((i_channel % in_channel) * size + i_col) * size + i_row];
}
}