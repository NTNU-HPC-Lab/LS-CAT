#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_bias_fc(float *preact, float *bias, const int n_channel)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int N = n_channel;

for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) {
preact[idx] += bias[idx];
}
}