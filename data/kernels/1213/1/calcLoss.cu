#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcLoss(float *err, float *output, unsigned int Y, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) {
err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
}
}