#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decat(float* input, float* output1, float* output2, float* output3, float* output4, const int size, const int out_channel1, const int out_channel2, const int out_channel3, const int out_channel4)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int in_channel = out_channel1 + out_channel2 + out_channel3 + out_channel4;  // # of channel of input
const int N = size * size;  // total elements per channel

if(pos < N){
for(int n = 0; n < in_channel; n++){
const int row = pos / size;
const int col = pos % size;
if(n < out_channel1){  // first output
output1[(n * size + col) * size + row] = input[(n * size + col) * size + row];
}
else if(n < out_channel1 + out_channel2){  // second output
output2[((n - out_channel1) * size + col) * size + row] = input[(n * size + col) * size + row];
}
else if(n < out_channel1 + out_channel2 + out_channel3){  // third output
output3[((n - out_channel1 - out_channel2) * size + col) * size + row] = input[(n * size + col) * size + row];
}
else{  // last output
output4[((n - out_channel1 - out_channel2 - out_channel3) * size + col) * size + row] = input[(n * size + col) * size + row];
}
}
}
}