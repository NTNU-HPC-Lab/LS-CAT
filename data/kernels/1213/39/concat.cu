#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void concat(float* input1, float* input2, float* input3, float* input4, size_t num1, size_t num2, size_t num3, size_t num4, size_t maxNum, float* output, const int numPerBatch)
{
size_t i = blockDim.x * blockIdx.x + threadIdx.x;

for(;i < maxNum; i += size_t(blockDim.x * gridDim.x)){
size_t batchIdx = i / numPerBatch; // which batch this thread is working in
const int batchOffset = i - batchIdx * numPerBatch; // offset of current thread in current batch

if(batchOffset < num1){  // first input
output[i] = input1[batchOffset + batchIdx * num1];
}
else if(batchOffset < (num1 + num2)){  // second input
output[i] = input2[(batchOffset - num1) + batchIdx * num2];
}
else if(batchOffset < (num1 + num2 + num3)){  // third input
output[i] = input3[(batchOffset - (num1 + num2)) + batchIdx * num3];
}
else{  // fourth input
output[i] = input4[(batchOffset - (num1 + num2 + num3)) + batchIdx * num4];
}
}
}