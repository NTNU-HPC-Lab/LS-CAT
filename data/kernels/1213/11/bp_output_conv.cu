#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_output_conv(float *d_output, float *weight, float *nd_preact, const int size, const int kernel_size, const int n_size, const int in_channel, const int out_channel, bool CONV, bool SAME)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int totalPos = blockDim.x * gridDim.x;

const int N = kernel_size * kernel_size * size * size * in_channel * out_channel;
const int weight_channel = out_channel * in_channel;
const int padding = (kernel_size - 1) / 2;   // must be int

for (int n = N * pos / totalPos; n < N * (pos+1) / totalPos; ++n) {
int idx = n;
int bpinput_row;
int bpinput_col;
const int i_channel = ((idx /= 1	) % weight_channel);
const int i_kernel_row = ((idx /= weight_channel) % kernel_size);
const int i_kernel_col = ((idx /= kernel_size) % kernel_size);
const int i_row = ((idx /= kernel_size	) % size);
const int i_col = ((idx /= size) % size);

if (SAME){ // SAME padding scheme implemented
bpinput_row = i_kernel_row + i_row - padding;
bpinput_col = i_kernel_col + i_col - padding;
}
else{
bpinput_row = i_kernel_row + i_row - 2 * padding;
bpinput_col = i_kernel_col + i_col - 2 * padding;
}

if(bpinput_row >= 0 && bpinput_row < n_size && bpinput_col >=0 && bpinput_col < n_size){
atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row],
weight[(i_channel * kernel_size + (kernel_size - 1 - i_kernel_col)) * kernel_size + kernel_size - 1 - i_kernel_row]
* nd_preact[((i_channel % out_channel) * n_size + bpinput_col) * n_size + bpinput_row]);
}
}
}