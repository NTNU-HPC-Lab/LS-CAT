#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixSubtract(const float* A_elements, const float* B_elements,  float* C_elements, const int size)
{
int thread = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;


for(int i = thread; i < size; i += stride)
//Modifying array of elements of Matrix C
C_elements[i] = A_elements[i] - B_elements[i];
}