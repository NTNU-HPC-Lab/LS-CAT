#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(float *VecA, float *VecB, float *VecC, int size)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < size)
VecC[i] = VecA[i] + VecB[i];
}