#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMultiply(const float* A_elements, const float* B_elements,  float* C_elements, const int X, const int Y, const int Z)
{
int baseMatrixRow = blockIdx.y * blockDim.y + threadIdx.y;
int baseMatrixCol = blockIdx.x * blockDim.x + threadIdx.x;

int strideX = blockDim.x * gridDim.x;
int strideY = blockDim.y * gridDim.y;

__shared__ float As[TILE_SIZE][TILE_SIZE];
__shared__ float Bs[TILE_SIZE][TILE_SIZE];

for (int iterY = 0; iterY < (Y + strideY - 1) / strideY; iterY++)
{
for (int iterX = 0; iterX < (X + strideX - 1)/ strideX; iterX++)
{
int matrixRow = baseMatrixRow + strideY * (iterY);
int matrixCol = baseMatrixCol + strideX * (iterX);

int blockRow = threadIdx.y;
int blockCol = threadIdx.x;

float Cvalue = 0;

for (int i = 0; i < ((X + TILE_SIZE - 1) / TILE_SIZE); ++i)
{

if((blockCol + i*TILE_SIZE) < X && matrixRow < Y)
As[blockRow][blockCol] = A_elements[matrixRow * X + blockCol + i*TILE_SIZE];
else
As[blockRow][blockCol] = 0;

if((blockRow + i*TILE_SIZE) < X && matrixCol < Z)
Bs[blockRow][blockCol] = B_elements[(blockRow + i*TILE_SIZE) * Z + matrixCol];
else
Bs[blockRow][blockCol] = 0;

//Synchronize threads
__syncthreads();

for (int j = 0; j < TILE_SIZE; ++j)
{
Cvalue += As[blockRow][j] * Bs[j][blockCol];
}

__syncthreads();
}
if (matrixRow < Y && matrixCol < Z) //Saving Final result into Matrix C
{
C_elements[matrixRow * Z + matrixCol] = Cvalue;
}
}
}
}