#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixTranspose(const float *A_elements, float *B_elements, const int A_width, const int A_height)
{
int strideRow = blockDim.y * gridDim.y;
int strideCol = blockDim.x * gridDim.x;

for(int row = blockIdx.y * blockDim.y + threadIdx.y; row < A_width; row += strideRow)
for(int col = blockIdx.x * blockDim.x + threadIdx.x; col < A_height; col += strideCol)
{
B_elements[row * A_height + col] = A_elements[col * A_width + row];
}
}