#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
int Row = blockIdx.y * blockDim.y + threadIdx.y;

int Col = blockIdx.x * blockDim.x + threadIdx.x;

if((Row < Width) && (Col < Width))
{
float Pvalue = 0;
for(int k = 0; k < Width; ++k)
{
Pvalue += M[Row*Width+k]*N[k*Width+Col];
}
P[Row*Width+Col] = Pvalue;
}
}