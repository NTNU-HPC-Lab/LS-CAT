#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexSelect_contiguous( float *tensor, float *src, long stride, float *index, long idxSize)
{
// In the typical case, each block of 128 threads handles a 4x128
// section of the output with each warp handling a single 1x128 row.
// The outer loops handle inputs larger than 4*65535 or strides larger
// than 128*65535.
const int VT = 4;
const int WARP_SIZE = 32;
const int MAX_DIM_SIZE = 65535;

for (int idx = blockIdx.x * blockDim.y + threadIdx.y; idx < idxSize; idx += blockDim.y * MAX_DIM_SIZE) {
for (int startIdx = threadIdx.x + blockIdx.y * VT*WARP_SIZE; startIdx < stride; startIdx += VT*WARP_SIZE*MAX_DIM_SIZE) {
const long srcIdx = ((long) index[idx] - 1) * stride;
const long targetIdx = idx * stride;

#pragma unroll
for (int i = 0; i < VT; i++) {
const int featureIdx = startIdx + i * WARP_SIZE;
if (featureIdx < stride) {
tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
}
}
}
}
}