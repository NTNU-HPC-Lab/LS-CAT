#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitonic_sort(int* arrayIn, int* arrayOut, int arrayLen, int chunkSize){
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < arrayLen) {
int myValue = arrayIn[idx];
int chunkStart = (idx / chunkSize) * chunkSize;
int chunkMid = chunkStart + (chunkSize / 2);
int partnerIndex = chunkSize - (idx - chunkStart) - 1 + chunkStart;
if (partnerIndex < arrayLen) {
int partnerValue = arrayIn[partnerIndex];
int min = (myValue <= partnerValue) ? myValue:partnerValue;
int max = (myValue > partnerValue) ? myValue:partnerValue;
myValue = (idx < chunkMid) ? min:max;
}
arrayOut[idx] = myValue;
}
}