#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void brickSort(int* array, int arrayLen, int p) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= arrayLen - 1)
return;
if ((p % 2 == 0) && (idx % 2 == 1))
return;
if ((p % 2 == 1) && (idx % 2 == 0))
return;
if (array[idx] > array[idx + 1]) {
int tmp = array[idx + 1];
array[idx + 1] = array[idx];
array[idx] = tmp;
}
}