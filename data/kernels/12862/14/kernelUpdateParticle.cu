#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateParticle(double *positions, double *velocities, double *pBests, double *gBest, int particlesCount, int dimensionsCount, double r1, double r2)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if(i >= particlesCount * dimensionsCount)
return;

velocities[i] = d_OMEGA * velocities[i] + r1 * (pBests[i] - positions[i])
+ r2 * (gBest[i % dimensionsCount] - positions[i]);

// Update posisi particle
positions[i] += velocities[i];
}