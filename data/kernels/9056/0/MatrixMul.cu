#include "hip/hip_runtime.h"
#include "includes.h"
#define THREADS_PER_BLOCK 256






__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{



int COL = threadIdx.x + blockIdx.x * blockDim.x;
int ROW = threadIdx.y + blockIdx.y * blockDim.y;



if (ROW < WIDTH && COL < WIDTH) {
for (int i = 0; i < WIDTH; i++) {
Pd[ROW * WIDTH + COL] += Md[ROW * WIDTH + i] * Nd [i * WIDTH + COL];
}
}

}