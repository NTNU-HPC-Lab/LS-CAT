#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multi(float *a, float *b, float *c, int width) {
__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

float result = 0;

for (int p = 0; p < width/TILE_WIDTH; p++)
{
s_a[threadIdx.y][threadIdx.x] = a[row*width + (p*TILE_WIDTH + threadIdx.x)];
s_b[threadIdx.y][threadIdx.x] = b[(p*TILE_WIDTH + threadIdx.y)*width + col];

__syncthreads();

for (int i = 0; i < TILE_WIDTH; i++)
{
result += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
}

__syncthreads();
}

c[row * width + col] = result;
}