#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{



int COL = threadIdx.x + blockIdx.x * blockDim.x;
int ROW = threadIdx.y + blockIdx.y * blockDim.y;



if (ROW < WIDTH && COL < WIDTH) {
for (int i = 0; i < WIDTH; i++) {
Pd[ROW * WIDTH + COL] += Md[ROW * WIDTH + i] * Nd [i * WIDTH + COL];
}
}

}