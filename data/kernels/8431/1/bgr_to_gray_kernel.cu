#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
// 2D Index of current thread
const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

// Only valid threads perform memory I/O
if ((xIndex < width) && (yIndex < height))
{
//Location of colored pixel in input
const int color_tid = yIndex * colorWidthStep + (3 * xIndex);

//Location of gray pixel in output
const int gray_tid = yIndex * grayWidthStep + xIndex;

const unsigned char blue = input[color_tid];
const unsigned char green = input[color_tid + 1];
const unsigned char red = input[color_tid + 2];

// The standard NTSC conversion formula that is used for calculating the effective luminance of a pixel (https://en.wikipedia.org/wiki/Grayscale#Luma_coding_in_video_systems)
const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

// Alternatively, use an average
//const float gray = (red + green + blue) / 3.f;

output[gray_tid] = static_cast<unsigned char>(gray);
}
}