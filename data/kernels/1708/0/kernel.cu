#include "hip/hip_runtime.h"
#include "includes.h"

/****************************************************************************
* An experiment with cuda kernel invocation parameters. 2x3x4 threads on
* one block should yield 24 kernel invocations.
*
* Compile with:
*   nvcc -o cupass cupass.cu
*
* Dr Kevan Buckley, University of Wolverhampton, January 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt){
char plain_password1[] ="AA1111";
char plain_password2[] ="AA1112";
char plain_password3[] ="AA1113";
char plain_password4[] ="AA1114";

char *q = attempt;
char *w = attempt;
char *e = attempt;
char *r = attempt;
char *pp1 = plain_password1;
char *pp2 = plain_password2;
char *pp3 = plain_password3;
char *pp4 = plain_password4;

while(*q ==*pp1){
if(*q == '\0')
{
printf("password:%s\n", plain_password1);
break;
}
q++;
pp1++;
}
while(*w ==*pp2){
if(*w == '\0')
{
printf("password:%s\n", plain_password2);
break;
}
w++;
pp2++;
}
while(*e ==*pp3){
if(*e == '\0')
{
printf("password:%s\n", plain_password3);
break;
}
e++;
pp3++;
}
while(*r ==*pp4){
if(*r == '\0')
{
printf("password: %s\n", plain_password4);
return 1;
}
r++;
pp4++;
}
return 0;
}
__global__ void kernel(){
char i1, i2, i3, i4;

char password[7];
password[6] ='\0';

int i = blockIdx.x +65;
int j = threadIdx.x+65;
char firstMatch =i;
char secondMatch =j;

password[0] =firstMatch;
password[1] =secondMatch;
for(i1='0'; i1<='9'; i1++){
for(i2='0'; i2<='9'; i2++){
for(i3='0'; i3<='9'; i3++){
for(i4='0'; i4<='9'; i4++){
password[2] =i1;
password[3] =i2;
password[4] =i3;
password[5] =i4;
if(is_a_match(password)){
}
else{
//printf("tried: %s\n",password);
}
}
}
}
}
}