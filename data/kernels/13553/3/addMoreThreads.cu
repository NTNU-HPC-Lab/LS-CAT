#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMoreThreads(int n, float *x, float *y)
{
// Let the kernel calculate which part of the input signal to play with
int index = threadIdx.x;
int stride = blockDim.x;

// Just did this to keep the syntax similar to the previous example
for (int i = index; i < n; i += stride)
y[i] = x[i] + y[i];
}