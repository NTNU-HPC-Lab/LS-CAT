#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addGridThreads(int n, float *x, float *y)
{
// Let the kernel calculate which part of the input signal to play with, but
// now also include the grid information
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride)
y[i] = x[i] + y[i];
}