#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_2d_mul_float_gpu(float *A, float *B, float *C, int num_rows_A, int num_cols_A, int num_cols_B) {
// Create shared variables (Available to all threads on the same block)
__shared__ float A_tile[N_THREADS][N_THREADS];
__shared__ float B_tile[N_THREADS][N_THREADS];
// Block index
int bx = blockIdx.x; int by = blockIdx.y;
// Thread index
int tx = threadIdx.x; int ty = threadIdx.y;

// Index of the first sub-matrix of A processed by the block
int aBegin = num_cols_A * N_THREADS * by;
// Index of the last sub-matrix of A processed by the block
int aEnd   = aBegin + num_cols_A - 1;
// Index of the first sub-matrix of B processed by the block
int bBegin = N_THREADS * bx;
int bStep  = N_THREADS * num_cols_B;
int aStep  = N_THREADS;

float sum = 0;

for (int a = aBegin, b = bBegin;a <= aEnd;a += aStep, b += bStep) {
A_tile[ty][tx] = A[a + num_cols_A * ty + tx];
B_tile[tx][ty] = B[b + num_cols_B * tx + ty];

// Synchronize to make sure the matrices are loaded
__syncthreads();

for (int k = 0; k < N_THREADS; ++k)
sum += A_tile[ty][k] * B_tile[k][tx];

// Wait other threads to finish their sub-matrices
__syncthreads();
}

// Write the block sub-matrix to device memory;
// each thread writes one element
int c = num_cols_B * N_THREADS * by + N_THREADS * bx;
C[c + num_cols_B * ty + tx] = sum;

}