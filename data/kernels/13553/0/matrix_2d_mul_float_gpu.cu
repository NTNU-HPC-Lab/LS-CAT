#include "hip/hip_runtime.h"
#include "includes.h"
/*
Now we make the matrix much bigger
g++ -pg seq_matrix_big_mul.c -o seq_matrix_big_mul
*/

#define N_THREADS 32

int num_rows_A = 2000; int num_rows_B = 2000; int num_rows_C = 2000;
int num_cols_A = 2000; int num_cols_B = 600; int num_cols_C = 600;
//int num_rows_A = 64; int num_rows_B = 64; int num_rows_C = 64;
//int num_cols_A = 64; int num_cols_B = 64; int num_cols_C = 64;

// I'm forcing a malloc because I want to add the malloc time on the game
float *A = (float*) malloc(sizeof(float) * num_rows_A * num_cols_A);
float *B = (float*) malloc(sizeof(float) * num_rows_B * num_cols_B);
float *C = (float*) malloc(sizeof(float) * num_rows_C * num_cols_C);
float *C_ref = (float*) malloc(sizeof(float) * num_rows_C * num_cols_C);


__global__ void matrix_2d_mul_float_gpu(float *A, float *B, float *C, int num_rows_A, int num_cols_A, int num_cols_B) {
// Same code for all 2d kernel
int i = blockIdx.y * blockDim.y + threadIdx.y;
int k = blockIdx.x * blockDim.x + threadIdx.x;
if (i > num_rows_A || k > num_cols_B) return;

float sum = 0;

for (int j=0; j<num_cols_A; j++){
// A[i][j] == A[i*num_cols_A+j]
// B[j][k] == B[j*num_cols_B+k]
//sum += A[i][j]*B[j][k];
sum += A[i*num_cols_A+j]*B[j*num_cols_B+k];
}

C[i*num_cols_B+k]=sum;
}