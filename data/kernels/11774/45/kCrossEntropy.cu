#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
target[i] = -mat[i] * __logf(p[i] + tiny);
}
}