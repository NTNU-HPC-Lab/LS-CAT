#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
dest[i] = alpha;
}
}