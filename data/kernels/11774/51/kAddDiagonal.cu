#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < width; i += numThreads) {
tgtMat[width*i + i] = mat[width*i + i] + vec[i];
}
}