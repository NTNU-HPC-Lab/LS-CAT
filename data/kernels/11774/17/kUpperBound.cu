#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
target[i] = mat1[i] > mat2[i] ? mat2[i] : mat1[i];
}
}