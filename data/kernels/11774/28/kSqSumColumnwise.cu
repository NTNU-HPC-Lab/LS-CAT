#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
__shared__ float sum_vals[32];
float cur_sum = 0;

for (unsigned int i = threadIdx.x; i < height; i += 32) {
cur_sum += mat[blockIdx.x * height + i] * mat[blockIdx.x * height + i];
}

sum_vals[threadIdx.x] = cur_sum;

__syncthreads();

if (threadIdx.x == 0) {
cur_sum = 0;

for (unsigned int i = 0; i < 32; i++)
cur_sum += sum_vals[i];

target[blockIdx.x] = cur_sum;
}
}