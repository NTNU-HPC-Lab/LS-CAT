#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = a[i] * (1.0 + b[i]) * (1.0 - b[i]) * 0.5;
}
}