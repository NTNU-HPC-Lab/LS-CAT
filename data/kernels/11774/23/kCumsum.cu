#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCumsum(float *mat, float *target, float *temp, unsigned int height) {
// extern  __shared__  float temp[];// allocated on invocation
const int thid = threadIdx.x;
if (2*thid < height) {
const int super_offset = blockIdx.x * height;
target += super_offset;
mat += super_offset;
temp += super_offset;
int offset = 1;
//float s = 0.0;
temp[2*thid]   = mat[2*thid]; // load input into shared memory
temp[2*thid+1] = mat[2*thid+1];
for (int d = height>>1; d > 0; d >>= 1) {// build sum in place up the tree
__syncthreads();
if (thid < d) {
int ai = offset*(2*thid+1)-1;
int bi = offset*(2*thid+2)-1;
temp[bi] += temp[ai];
} else if (thid == d && thid % 2 == 1) {
//int bi = offset*(2*thid+2)-1;
//temp[bi] += temp[ai];

}

offset *= 2;
}
if (thid == 0) { temp[height - 1] = 0; } // clear the last element
for (int d = 1; d < height; d *= 2)  { // traverse down tree & build scan
offset >>= 1;
__syncthreads();
if (thid < d) {
int ai = offset*(2*thid+1)-1;
int bi = offset*(2*thid+2)-1;
float t   = temp[ai];
temp[ai]  = temp[bi];
temp[bi] += t;
}
}
__syncthreads();
// write results to device memory
//  if (thid == -1) {
//    target[0]   = temp[1];
//    target[height-1] = s;
//  } else {
target[2*thid]   = temp[2*thid];
target[2*thid+1] = temp[2*thid+1];
//  }
}

}