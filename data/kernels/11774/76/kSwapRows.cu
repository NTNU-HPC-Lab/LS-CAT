#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSwapRows(float* source, float* target, float* indices1, float* indices2, int nRowIs, int nCols, int nRows){
__shared__ int sourceRowIndices[32], targetRowIndices[32];
const int startRowI = blockIdx.x * 32;
const int tid = threadIdx.x;
const int localNRowIs = min(32, nRowIs-startRowI);

// cooperatively load 32 row indices
if (tid < localNRowIs){
sourceRowIndices[tid] = int(indices1[startRowI + tid]);
targetRowIndices[tid] = int(indices2[startRowI + tid]);
if (sourceRowIndices[tid]<0)
sourceRowIndices[tid] += nRows;
if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nRows)
sourceRowIndices[tid] = -1;
if (targetRowIndices[tid]<0)
targetRowIndices[tid] += nRows;
if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nRows)
targetRowIndices[tid] = -1;
}
__syncthreads();

// copy 32 rows
for (int i=0; i<localNRowIs; i++){
const int sourceRowI = sourceRowIndices[i], targetRowI = targetRowIndices[i];
for (int colI=tid; colI<nCols; colI+=32) {
const float temp1 = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
const float temp2 = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : target[targetRowI * nCols + colI];
if (sourceRowI != -1)
source[sourceRowI * nCols + colI] =  temp2;
if (targetRowI != -1)
target[targetRowI * nCols + colI] = temp1;
}
}
}