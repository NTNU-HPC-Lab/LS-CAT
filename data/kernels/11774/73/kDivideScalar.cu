#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
dest[i] = mat[i] / alpha;
}
}