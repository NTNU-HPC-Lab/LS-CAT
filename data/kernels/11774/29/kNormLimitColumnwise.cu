#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kNormLimitColumnwise(float* mat, float* target, float norm, unsigned int width, unsigned int height) {
__shared__ float sum_vals[33];
float cur_sum = 0;
for (unsigned int i = threadIdx.x; i < height; i += 32) {
cur_sum += mat[blockIdx.x * height + i] * mat[blockIdx.x * height + i];
}
sum_vals[threadIdx.x] = cur_sum;
__syncthreads();
if (threadIdx.x == 0) {
cur_sum = 0;
for (unsigned int i = 0; i < 32; i++)
cur_sum += sum_vals[i];
sum_vals[32] = norm > cur_sum ? 1 : norm / sqrt(cur_sum);
}
float scale = sum_vals[32];
for (unsigned int i = threadIdx.x; i < height; i += 32) {
target[blockIdx.x * height + i] = mat[blockIdx.x * height + i] * scale;
}
__syncthreads();
}