#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
target[i] = __logf(mat[i] + tiny);
}
}