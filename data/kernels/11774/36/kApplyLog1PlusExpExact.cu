#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyLog1PlusExpExact(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
float mat_i;

for (unsigned int i = idx; i < len; i += numThreads) {
mat_i = mat[i];
if (mat_i > 0)
target[i] = (log(1 + exp(-mat_i)) + mat_i);
else
target[i] = log(1 + exp(mat_i));
}
}