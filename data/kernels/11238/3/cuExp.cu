#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

__global__ void cuExp(int n, float *a, float *result)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
result[i] = expf(a[i]);
}

}