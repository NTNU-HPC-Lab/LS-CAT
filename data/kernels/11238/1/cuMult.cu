#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

__global__ void cuMult(int n, float *a, float *b, float *result)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
result[i] = a[i] * b[i];
}

}