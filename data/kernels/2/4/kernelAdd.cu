#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelAdd(float const *in1, float const *in2, int _w, int _h, float *out) {

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= _w || y >= _h)
return;

out[y*_w + x] = __fsqrt_rn(__fadd_rn(__fmul_rn(in1[y*_w + x],in1[y*_w + x]), __fmul_rn(in2[y*_w + x],in2[y*_w + x])));
}