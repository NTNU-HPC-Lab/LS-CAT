#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelScharrY( float const *in, int _w, int _h, float *out )
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= _w || y >= _h)
return;

// Pattern  // Indexes:
//  -3 0  3 // a1 b1 c1
// -10 0 10 // a2 b2 c2
//  -3 0  3 // a3 b3 c3

int a = max(y-1,0);
int c = min((y+1),_h -1);

int a1, a2, a3,
c1, c2, c3;

int i1 = max(x-1, 0);
int i3 = min(x+1, _w-1);

a1 = a*_w + i1;
a2 = a*_w + x;
a3 = a*_w + i3;

c1 = c*_w + i1;
c2 = c*_w + x;
c3 = c*_w + i3;

out[y*_w+x] = __fdividef(3.0*(- in[a1] -in[a3] +in[c1] +in[c3])
+ 10.0*(in[c2] -in[a2]), 20.0);

//  out[y*_w+x] = -3.0*in[a1] -10.0*in[a2] -3.0*in[a3] + 3.0*in[c1] + 10.0*in[c2] + 3.0*in[c3];
}