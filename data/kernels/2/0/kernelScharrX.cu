#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelScharrX( float const *in, int _w, int _h, float *out) {
// Pattern : // Indexes :
// -3 -10 -3 // a1 b1 c1
//  0   0  0 // a2 b2 c2
//  3  10  3 // a3 b3 c3

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= _w || y >= _h)
return;

int a = max(y-1,0);
int b = y;
int c = min((y+1),_h -1);

int a1, a3,
b1, b3,
c1, c3;

int i1 = max(x-1, 0);
int i3 = min(x+1, _w-1);

a1 = a*_w + i1;
a3 = a*_w + i3;

b1 = b*_w + i1;
b3 = b*_w + i3;

c1 = c*_w + i1;
c3 = c*_w + i3;

out[y*_w+x] = __fdividef(3.0 * (-in[a1]  -in[c1] + in[a3] + in[c3])
+ 10.0 * (in[b3] -in[b1]), 20.0);

//  out[y*_w+x] = -3.0*in[a1] -10.0*in[b1] -3.0*in[c1] + 3.0*in[a3] + 10.0*in[b3] + 3.0*in[c3];
}