#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelSmoothY(float const * in, int w, int h, float * out)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= w || y >= h)
return;

int a = y-2;
int b = y-1;
int c = y;
int d = y+1;
int e = y+2;

if(a < 0) a = 0;
if(b < 0) b = 0;
if(d >= h) d = h-1;
if(e >= h) e = h-1;

out[y*w+x] = 0.0625f*in[a*w+x] + 0.25f*in[b*w+x] + 0.375f*in[c*w+x] + 0.25f*in[d*w+x] + 0.0625f*in[e*w+x];
}