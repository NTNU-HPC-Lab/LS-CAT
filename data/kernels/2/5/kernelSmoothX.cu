#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelSmoothX(float *in, int w, int h, float *out)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= w || y >= h)
return;

int idx = y*w;

int a = x-2;
int b = x-1;
int c = x;
int d = x+1;
int e = x+2;

if(a < 0) a = 0;
if(b < 0) b = 0;
if(d >= w) d = w-1;
if(e >= w) e = w-1;

out[y*w+x] = 0.0625f*in[idx+a] + 0.25f*in[idx+b] + 0.375f*in[idx+c] + 0.25f*in[idx+d] + 0.0625f*in[idx+e];
}