#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelSobelY( float const *in, int _w, int _h, float *out)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= _w || y >= _h)
return;

// Pattern  // Indexes:
//  -1 0 1 // a1 b1 c1
// - 2 0 2 // a2 b2 c2
//  -1 0 1 // a3 b3 c3

int a = max(y-1,0);
int c = min((y+1),_h -1);

int a1, a2, a3,
c1, c2, c3;

int i1 = max(x-1, 0);
int i3 = min(x+1, _w-1);

a1 = a*_w + i1;
a2 = a*_w + x;
a3 = a*_w + i3;

c1 = c*_w + i1;
c2 = c*_w + x;
c3 = c*_w + i3;

out[y*_w + x] = __fdividef(-1.0*in[a1] -2.0*in[a2] -1.0*in[a3] + 1.0*in[c1] + 2.0*in[c2] + 1.0*in[c3], 4.0);
}