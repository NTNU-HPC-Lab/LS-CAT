#include "hip/hip_runtime.h"
#include "includes.h"



// Constant values on device
// /!\ undefined in host code, just in kernels /!\ __device__
#define MAX_WEIGHT_VALUES 50
#define MIN_DET FLT_EPSILON

__constant__ __device__ int   LK_iteration;
__constant__ __device__ int   LK_patch;
__constant__ __device__ int   LK_points;
__constant__ __device__ int   LK_height;
__constant__ __device__ int   LK_width;
__constant__ __device__ int   LK_pyr_w;
__constant__ __device__ int   LK_pyr_h;
__constant__ __device__ int   LK_pyr_level;
__constant__ __device__ int   LK_width_offset;
__constant__ __device__ char  LK_init_guess;
__constant__ __device__ float LK_scaling;
__constant__ __device__ float LK_threshold;
__constant__ __device__ float LK_Weight[MAX_WEIGHT_VALUES];
__constant__ __device__ int   LK_win_size;

// Texture buffer is used for each image for on-the-fly interpolation
texture <float, 2, hipReadModeElementType> texRef_pyramid_prev;
texture <float, 2, hipReadModeElementType> texRef_pyramid_cur;

// Image pyramids -> texture buffers
texture <float, 2, hipReadModeElementType> gpu_textr_pict_0;   // pictures > texture space
texture <float, 2, hipReadModeElementType> gpu_textr_pict_1;

texture <float, 2, hipReadModeElementType> gpu_textr_deriv_x;  // gradients > texture space
texture <float, 2, hipReadModeElementType> gpu_textr_deriv_y;

// Convert RGB Picture to grey/float

// Convert Grey uchar picture to float

// Downsample picture to build pyramid lower level (naive implementation..)


// Kernel to compute the tracking

// Kernel to compute the tracking
__global__ void convertRGBToGrey(unsigned char *d_in, float *d_out, int N)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if(idx < N)
{
d_out[idx] = d_in[idx*3]*0.1144f
+ d_in[idx*3+1]*0.5867f
+ d_in[idx*3+2]*0.2989f;
}
}