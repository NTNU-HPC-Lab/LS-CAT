#include "hip/hip_runtime.h"
#include "includes.h"



// Possible weight coefficients for tracking cost evaluation :
// Gaussian discretisation
/*
*       1  4  6  4  1
*       4 16 24 16  4
*       6 24 36 24  6
*       4 16 24 16  4
*       1  4  6  4  1
*/


// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Scharr operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..

// Compute spatial derivatives using Sobel operator - Naive implementation..



// Low pass gaussian-like filtering before subsampling

// Low pass gaussian-like filtering before subsampling

/*
// Upsample a picture using the "magic" kernel
*/
__global__ void kernelMagicUpsampleX(float *in, int _w, int _h, float *out) {
// Coefficients : 1/4, 3/4, 3/4, 1/4 in each direction (doubles the size of the picture)

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if(x >= _w || y >= _h)
return;

// Duplicate the points at the same place (?)
out[y*2*_w + 2*x] = in[y*_w+x];


if ((x < (_w-2)) && (x > 1))
out[y*2*_w + 2*x + 1] = __fdividef(3.0*(in[y*_w+x] + in[y*_w + x + 1]) + in[y*_w+x -1] + in[y*_w+x +2] , 8.0);

}