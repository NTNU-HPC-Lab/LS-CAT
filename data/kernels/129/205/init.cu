#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int *vector, int N, int val)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;

if (i < N) {
vector[i] = val;
}
}