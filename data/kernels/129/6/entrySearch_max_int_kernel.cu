#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void entrySearch_max_int_kernel(int *g_iarr, int *g_maxarr, int size)
{
// create shared memory
extern __shared__ int sarr_int[];

// load shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

if(i + blockDim.x < size) {
if(g_iarr[i] > g_iarr[i + blockDim.x]) {
sarr_int[tid] = g_iarr[i];
} else {
sarr_int[tid] = g_iarr[i + blockDim.x];
}
} else if (i < size) {
sarr_int[tid] = g_iarr[i];
} else {
sarr_int[tid] = INT_MIN;
}

__syncthreads();

// do comparison in shared mem
for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
if(tid < s) {
if(sarr_int[tid] < sarr_int[tid + s]) {
sarr_int[tid] = sarr_int[tid + s];
}
}
__syncthreads();
}

// write result for this block to global mem
if(tid == 0) {
g_maxarr[blockIdx.x] = sarr_int[0];
}
}