#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
int idx = blockDim.x * blockIdx.x + threadIdx.x;
if(idx >= sparseSize)
return;
if(sparseIndices[idx] >= -offset && sparseIndices[idx] + offset < denseSize)
denseData[sparseIndices[idx] + offset] += sparseData[idx];
}