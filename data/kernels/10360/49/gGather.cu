#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
int idx = blockDim.x * blockIdx.x + threadIdx.x;
if(idx >= sparseSize)
return;
if(sparseIndices[idx] >= -offset && sparseIndices[idx] + offset < denseSize)
sparseData[idx] = denseData[sparseIndices[idx] + offset];
}