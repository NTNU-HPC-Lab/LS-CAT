#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ZeroMeanImpl(float* solutions, int rowSize, int matCount) {

const int matricesPerBlock = BLOCK_SIZE / rowSize;

const int matrixIdx = blockIdx.x * matricesPerBlock + threadIdx.x / rowSize;
const int tid = threadIdx.x;
const int col = threadIdx.x & (rowSize - 1);
const int inBlockOffset = threadIdx.x / rowSize;

__shared__ double beta[BLOCK_SIZE];
__shared__ double line[BLOCK_SIZE];

if (matrixIdx >= matCount) {
return;
}

solutions += matrixIdx * rowSize;
beta[tid] = col != (rowSize - 1) ? solutions[col] : 0;
line[tid] = beta[tid];
__syncthreads();

for (int s = rowSize >> 1; s > 0; s >>= 1) {
if (col < s) {
line[tid] += line[tid + s];
}
__syncthreads();
}

beta[tid] -= line[rowSize * inBlockOffset] / rowSize;
solutions[col] = beta[tid];
}