#include "hip/hip_runtime.h"
#include "includes.h"
// In CUDA we trust.

// When compiling, use -std=c++11 or higher.



__global__ void histogramSimple(int* d_out, const int* d_in, const int BINS_COUNT) {
int tid = threadIdx.x + blockDim.x * blockIdx.x;
atomicAdd(&(d_out[d_in[tid] % BINS_COUNT]), 1);
}