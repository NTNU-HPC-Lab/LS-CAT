#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogramSimple(int* d_out, const int* d_in, const int BINS_COUNT) {
int tid = threadIdx.x + blockDim.x * blockIdx.x;
atomicAdd(&(d_out[d_in[tid] % BINS_COUNT]), 1);
}