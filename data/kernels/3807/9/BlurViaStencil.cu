#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BlurViaStencil(float* d_out, float* d_in) {
const float kernel[3][3] = {0.04, 0.12, 0.04,
0.12, 0.36, 0.12,
0.04, 0.12, 0.04};
int rowID = blockIdx.x + 1;
int colID = threadIdx.x + 1;
int pos = rowID * (blockDim.x + 2) + colID;
d_out[pos] = d_in[pos - blockDim.x - 3] * kernel[0][0]
+ d_in[pos - blockDim.x - 2] * kernel[0][1]
+ d_in[pos - blockDim.x - 1] * kernel[0][2]
+ d_in[pos - 1] * kernel[1][0]
+ d_in[pos] * kernel[1][1]
+ d_in[pos + 1] * kernel[1][2]
+ d_in[pos + blockDim.x + 1] * kernel[2][0]
+ d_in[pos + blockDim.x + 2] * kernel[2][1]
+ d_in[pos + blockDim.x + 3] * kernel[2][2];
}