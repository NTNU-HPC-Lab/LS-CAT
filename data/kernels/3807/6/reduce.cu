#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(float* d_out, float* d_in) { // Parallel summation: steps = O(log(N)), work = O(N * log(N))
extern __shared__ float sdata[];

int globId = blockDim.x * blockIdx.x + threadIdx.x;
int tid = threadIdx.x;

sdata[tid] = d_in[globId];
__syncthreads();

int s = blockDim.x >> 1;
while (s > 0) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
s >>= 1;
}
if (tid == 0) {
d_out[blockIdx.x] = sdata[0];
}
}