#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Laplace(float* d_out, float* d_in) {
int rowID = blockIdx.x + 1;
int colID = threadIdx.x + 1;
int pos = rowID * (blockDim.x + 2) + colID;
d_out[pos] = (d_in[pos - 1] + d_in[pos + 1] +
d_in[pos - blockDim.x - 2] + d_in[pos + blockDim.x + 2]) /  4.;
}