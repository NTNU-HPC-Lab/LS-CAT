#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prescan(float* d_in, int nGlobe, int step, int upSweep) {
int tid = blockDim.x * blockIdx.x + threadIdx.x;
int from = 2 * tid * (step + 1) + step;
int to = 2 * tid * (step + 1) + 2 * step + 1;
if (upSweep) {
d_in[to] += d_in[from];
} else {
int temp = d_in[to];
d_in[to] += d_in[from];
d_in[from] = temp;
}
}