#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void montecarlo(float* d_out, float __lowx, float __highx, float __lowy, float __highy, int __iters) {
__shared__ float lowx, highx, lowy, highy;
__shared__ int iters;

int tid = blockIdx.x * blockDim.x + threadIdx.x;

// let's fix the shared variables for all threads per block once (check the synchronization call).
if (threadIdx.x == 0) {
lowx = __lowx, highx = __highx, lowy = __lowy, highy = __highy;
iters = __iters;
}
__syncthreads();

hiprandState localState;
hiprand_init(tid, 0, 0, &localState);

int i;
float x, y, tempSum = 0.;
for (i = 0; i < iters; i ++) { // each thread calculates its own summation.
x = lowx + hiprand_uniform(&localState) * (highx - lowx);
y = lowy + hiprand_uniform(&localState) * (highy - lowy);
tempSum += exp(-x * x - y * y);
}
d_out[tid] = tempSum;
}