#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeHessianListS0(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22, double *point_gradients0, double *point_gradients1, double *point_gradients2, double *tmp_hessian, int valid_voxel_num)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int col = blockIdx.y;

if (col < 6) {
double *tmp_pg0 = point_gradients0 + col * valid_points_num;
double *tmp_pg1 = point_gradients1 + 6 * valid_points_num;
double *tmp_pg2 = point_gradients2 + 6 * valid_points_num;
double *tmp_h = tmp_hessian + col * valid_voxel_num;

for (int i = id; i < valid_points_num && col < 6; i += stride) {
int pid = valid_points[i];
double d_x = static_cast<double>(trans_x[pid]);
double d_y = static_cast<double>(trans_y[pid]);
double d_z = static_cast<double>(trans_z[pid]);

double pg0 = tmp_pg0[i];
double pg1 = tmp_pg1[i];
double pg2 = tmp_pg2[i];

for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
int vid = voxel_id[j];

tmp_h[j] = (d_x - centroid_x[vid]) * (icov00[vid] * pg0 + icov01[vid] * pg1 + icov02[vid] * pg2)
+ (d_y - centroid_y[vid]) * (icov10[vid] * pg0 + icov11[vid] * pg1 + icov12[vid] * pg2)
+ (d_z - centroid_z[vid]) * (icov20[vid] * pg0 + icov21[vid] * pg1 + icov22[vid] * pg2);
}
}
}
}