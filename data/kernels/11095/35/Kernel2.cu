#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel2(bool *g_graph_mask, bool *g_updating_graph_mask, bool *g_graph_visited, bool *g_over, int no_of_nodes) {
int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
if (tid < no_of_nodes && g_updating_graph_mask[tid]) {

g_graph_mask[tid] = true;
g_graph_visited[tid] = true;
*g_over = true;
g_updating_graph_mask[tid] = false;
}
}