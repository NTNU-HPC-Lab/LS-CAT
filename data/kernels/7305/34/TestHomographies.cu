#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TestHomographies(float *d_coord, float *d_homo, int *d_counts, int numPts, float thresh2)
{
__shared__ float homo[8*TESTHOMO_LOOPS];
__shared__ int cnts[TESTHOMO_TESTS*TESTHOMO_LOOPS];
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int idx = blockIdx.y*blockDim.y + tx;
const int numLoops = blockDim.y*gridDim.y;
if (ty<8 && tx<TESTHOMO_LOOPS)
homo[tx*8+ty] = d_homo[idx+ty*numLoops];
__syncthreads();
float a[8];
for (int i=0;i<8;i++)
a[i] = homo[ty*8+i];
int cnt = 0;
for (int i=tx;i<numPts;i+=TESTHOMO_TESTS) {
float x1 = d_coord[i+0*numPts];
float y1 = d_coord[i+1*numPts];
float x2 = d_coord[i+2*numPts];
float y2 = d_coord[i+3*numPts];
float nomx = __fmul_rz(a[0],x1) + __fmul_rz(a[1],y1) + a[2];
float nomy = __fmul_rz(a[3],x1) + __fmul_rz(a[4],y1) + a[5];
float deno = __fmul_rz(a[6],x1) + __fmul_rz(a[7],y1) + 1.0f;
float errx = __fmul_rz(x2,deno) - nomx;
float erry = __fmul_rz(y2,deno) - nomy;
float err2 = __fmul_rz(errx,errx) + __fmul_rz(erry,erry);
if (err2<__fmul_rz(thresh2,__fmul_rz(deno,deno)))
cnt ++;
}
int kty = TESTHOMO_TESTS*ty;
cnts[kty + tx] = cnt;
__syncthreads();
int len = TESTHOMO_TESTS/2;
while (len>0) {
if (tx<len)
cnts[kty + tx] += cnts[kty + tx + len];
len /= 2;
__syncthreads();
}
if (tx<TESTHOMO_LOOPS && ty==0)
d_counts[idx] = cnts[TESTHOMO_TESTS*tx];
__syncthreads();
}