#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(double *a, int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx<N) a[idx] = a[idx] * a[idx];
printf("idx = %d, a = %f\n", idx, a[idx]);
}