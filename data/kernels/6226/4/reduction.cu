#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(float *g_odata, float *g_idata)
{
// dynamically allocated shared memory

extern  __shared__  float temp[];

int tid = threadIdx.x + blockIdx.x*blockDim.x;
int k = threadIdx.x;

// first, each thread loads data into shared memory

temp[k] = g_idata[tid];

// next, we perform binary tree reduction
int d = blockDim.x; if (d % 2) temp[0] += temp[d - 1];
for (d >>= 1; d > 0; d >>= 1) {
__syncthreads();  // ensure previous step completed
if (k<d) { temp[k] += temp[k + d]; }
if (k == 0 && d % 2 == 1 && d != 1) { temp[0] += temp[d - 1]; }
//printf("middle result:d:%d  temp[%d]:%f\n",d,k,temp[k]);
}

// finally, first thread puts result into global memory

if (tid == blockIdx.x*blockDim.x) {
g_odata[blockIdx.x] = temp[0];
//printf("g[%d]:%f\n",blockIdx.x,g_odata[blockIdx.x]);
}
}