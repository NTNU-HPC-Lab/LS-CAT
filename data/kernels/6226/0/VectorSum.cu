#include "hip/hip_runtime.h"
#include "includes.h"


hipError_t sumWithCuda(int *c, const int *a, const int *b, unsigned int size);

//ÄÚºËº¯Êý

__global__ void VectorSum(int *result, const int *vector_a, const int *vector_b)
{
int i = threadIdx.x;
result[i] = vector_a[i] + vector_b[i];
printf("%d : call kernel function.\n", i);
}