#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void run_reduction(int *con, int *blockCon,int* ActiveList, int nActiveBlock, int* blockSizes)
{
int list_idx = blockIdx.x;
int tx = threadIdx.x;
int block_idx = ActiveList[list_idx];
int start = block_idx*blockDim.x * 2;
int blocksize = blockSizes[block_idx];
__shared__ int s_block_conv;
s_block_conv = 1;
__syncthreads();

if (tx < blocksize)
{
if (!con[start + tx])
s_block_conv = 0;
}
__syncthreads();

if(tx == 0)
{
blockCon[block_idx] = s_block_conv; // active list is negation of tile convergence (active = not converged)
}
}