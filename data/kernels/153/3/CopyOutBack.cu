#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float3 operator % (const float3 & v1, const float3 & v2)
{   return make_float3(v1.y*v2.z - v1.z*v2.y,
v1.z*v2.x - v1.x*v2.z,
v1.x*v2.y - v1.y*v2.x);}
__global__ void CopyOutBack(float4* d_tetT, float* d_vertT, int* d_vertMem, int* d_vertMemOutside, int* d_BlockSizes, int* d_ActiveList,int m_maxNumInVert,int m_maxNumTotalTets, int m_maxVertMappingInside, int m_maxVertMappingOutside)
{
int list_idx = blockIdx.y*gridDim.x + blockIdx.x;
// retrieve actual block index from the active list
int block_idx = d_ActiveList[list_idx];
int block_size = d_BlockSizes[block_idx];

////////////initialize shared memory//////////////////////////////////////////

int tx = threadIdx.x;
int tet_base = block_idx*m_maxNumTotalTets;
int vert_base = block_idx*m_maxNumInVert;
int tmpindex;

if(tx < block_size)
{
float T = d_vertT[vert_base + tx];

int j =0;
tmpindex = d_vertMem[block_idx*m_maxVertMappingInside*m_maxNumInVert +  tx * m_maxVertMappingInside + j];
while(j < m_maxVertMappingInside && (tmpindex - tet_base*4) > -1) // update gloal memory inside all the old to the min
{
int segment = tmpindex / 4;
int offset = tmpindex % 4;
switch(offset)
{
case 0:
d_tetT[segment].x = T;
break;
case 1:
d_tetT[segment].y = T;
break;
case 2:
d_tetT[segment].z = T;
break;
case 3:
d_tetT[segment].w = T;
break;
}
j++;
tmpindex = d_vertMem[block_idx*m_maxVertMappingInside*m_maxNumInVert +  tx * m_maxVertMappingInside + j];
}

for(int j = 0; (j < m_maxVertMappingOutside) && ((tmpindex = d_vertMemOutside[block_idx*m_maxVertMappingOutside*m_maxNumInVert +  tx * m_maxVertMappingOutside + j]) > -1 ); j++) // update gloal memory outside all the old to the min
{
int segment = tmpindex / 4;
int offset = tmpindex % 4;
switch(offset)
{
case 0:
d_tetT[segment].x = T;
break;
case 1:
d_tetT[segment].y = T;
break;
case 2:
d_tetT[segment].z = T;
break;
case 3:
d_tetT[segment].w = T;
break;
}
}
}
}