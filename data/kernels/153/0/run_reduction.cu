#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void run_reduction(bool *con, bool *blockCon,int* ActiveList, int nActiveBlock, int* blockSizes)
{
int list_idx = blockIdx.y*gridDim.x + blockIdx.x;
int maxblocksize = blockDim.x;
int tx = threadIdx.x;
int block_idx = ActiveList[list_idx];

int blocksize = blockSizes[block_idx];

__shared__ bool s_block_conv;


s_block_conv = true;
__syncthreads();

if(tx < blocksize)
{
if(!con[maxblocksize*block_idx+tx])
s_block_conv= false;
}
__syncthreads();

if(tx == 0)
{
blockCon[block_idx] = s_block_conv; // active list is negation of tile convergence (active = not converged)
}
}