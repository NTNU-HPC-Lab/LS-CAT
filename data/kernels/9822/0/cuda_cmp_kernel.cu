#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
int i = threadIdx.x+blockIdx.x*blockDim.x;
int cmp = i<n? aptr[i]<bptr[i]: 0;
if (__syncthreads_or(cmp)) *rptr=1;
}