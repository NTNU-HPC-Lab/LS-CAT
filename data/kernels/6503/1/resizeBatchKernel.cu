#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void resizeBatchKernel(const uint8_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *p_dst, int nDstWidth, int nDstHeight) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uchar3 rgb;
int nDstW = nDstWidth;
int nDstH = nDstHeight;
int yScale = nSrcHeight / nDstHeight;
int xScale = 3 * (nSrcPitch / nDstWidth);
if (tid < nDstW && tidd < nDstH) {
int j = tidd * yScale * nSrcPitch * 3;
int k = tid * xScale;
rgb.x = p_Src[j + k + 0];
rgb.y = p_Src[j + k + 1];
rgb.z = p_Src[j + k + 2];
k = tid * 3;
j = tidd * nDstWidth * 3;
p_dst[j + k + 0] = rgb.x;
p_dst[j + k + 1] = rgb.y;
p_dst[j + k + 2] = rgb.z;
}
}