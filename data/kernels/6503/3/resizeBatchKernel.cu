#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint16_t *p_dst, int nDstWidth, int nDstHeight) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uint4 pF;
int scale = nSrcHeight / nDstHeight;
if (scale == 4) {
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 6;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 32;

pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
pF.w = (uint32_t)p_Src[j + k + 6];

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);
y1 = (uint32_t)(pF.w & 0x000003FF);

pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
pF.z = (uint32_t)p_Src[j + k + 12];

y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
v1 = (uint32_t)(pF.z & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
y4 = (uint32_t)(pF.w & 0x000003FF);

pF.y = ((uint32_t)p_Src[j + k + 27] << 16);

y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

k = tid * 6;
j = tidd * nDstWidth;
p_dst[j + k + 0] = y0;
p_dst[j + k + 1] = y1;
p_dst[j + k + 2] = y2;
p_dst[j + k + 3] = y3;
p_dst[j + k + 4] = y4;
p_dst[j + k + 5] = y5;
k = tid * 3;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
p_dst[j + k + 0] = u0;
p_dst[j + k + 1] = u1;
p_dst[j + k + 2] = u2;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
p_dst[j + k + 0] = v0;
p_dst[j + k + 1] = v1;
p_dst[j + k + 2] = v2;
}
} else if (scale == 6) {
uint32_t v0, y0, u0, y1;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 2;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 16;
pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

k = tid * 2;
j = tidd * nDstWidth;
p_dst[j + k + 0] = y0;
p_dst[j + k + 1] = y1;
k = tid;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
p_dst[j + k + 0] = u0;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
p_dst[j + k + 0] = v0;
}
} else if (scale == 2) {
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 6;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 16;
pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);
y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
y2 = (uint32_t)(pF.w & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
pF.z = (uint32_t)p_Src[j + k + 12];
pF.w = (uint32_t)p_Src[j + k + 14];

y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
v2 = (uint32_t)(pF.z & 0x000003FF);
y5 = (uint32_t)(pF.w & 0x000003FF);

k = tid * 6;
j = tidd * nDstWidth;
p_dst[j + k + 0] = y0;
p_dst[j + k + 1] = y1;
p_dst[j + k + 2] = y2;
p_dst[j + k + 3] = y3;
p_dst[j + k + 4] = y4;
p_dst[j + k + 5] = y5;
k = tid * 3;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
p_dst[j + k + 0] = u0;
p_dst[j + k + 1] = u1;
p_dst[j + k + 2] = u2;
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
p_dst[j + k + 0] = v0;
p_dst[j + k + 1] = v1;
p_dst[j + k + 2] = v2;
}
}
}