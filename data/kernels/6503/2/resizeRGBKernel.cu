#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void resizeRGBKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *p_dst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
ushort3 rgb;
int nDstW = nDstWidth;
int nDstH = nDstHeight;
int yScale = nSrcHeight / nDstHeight;
int xScale = 3 * (nSrcPitch / nDstWidth);
if (tid < nDstW && tidd < nDstH) {
int j = tidd * yScale * nSrcPitch * 3;
int k = tid * xScale;
rgb.x = p_Src[j + k + 0];
rgb.y = p_Src[j + k + 1];
rgb.z = p_Src[j + k + 2];
k = tid * 3;
j = tidd * nDstWidth * 3;
p_dst[j + k + 0] = lookupTable_cuda[rgb.x];
p_dst[j + k + 1] = lookupTable_cuda[rgb.y];
p_dst[j + k + 2] = lookupTable_cuda[rgb.z];
}
}