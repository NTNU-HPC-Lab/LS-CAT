#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void rgbToV210Kernel(uint16_t *pSrc, uint16_t *pDst, int nSrcWidth, int nHeight) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
uint3 rgb;
uint4 pF;
int nDstW = nSrcWidth / 18;
int nDstH = nHeight;
if (tid < nDstW && tidd < nDstH) {
int k = tid * 18;
int j = tidd * nSrcWidth;
rgb.x = pSrc[j + k + 0];
rgb.y = pSrc[j + k + 1];
rgb.z = pSrc[j + k + 2];

y0 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;
u0 = (512000 - rgb.x * 169 - rgb.y * 332 + rgb.z * 500) / 1000;
v0 = (512000 + rgb.x * 500 - rgb.y * 419 - rgb.z * 81) / 1000;

rgb.x = pSrc[j + k + 3];
rgb.y = pSrc[j + k + 4];
rgb.z = pSrc[j + k + 5];

y1 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;
u1 = (512000 - rgb.x * 169 - rgb.y * 332 + rgb.z * 500) / 1000;
v1 = (512000 + rgb.x * 500 - rgb.y * 419 - rgb.z * 81) / 1000;

rgb.x = pSrc[j + k + 6];
rgb.y = pSrc[j + k + 7];
rgb.z = pSrc[j + k + 8];

y2 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;
u2 = (512000 - rgb.x * 169 - rgb.y * 332 + rgb.z * 500) / 1000;
v2 = (512000 + rgb.x * 500 - rgb.y * 419 - rgb.z * 81) / 1000;

rgb.x = pSrc[j + k + 9];
rgb.y = pSrc[j + k + 10];
rgb.z = pSrc[j + k + 11];

y3 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;

rgb.x = pSrc[j + k + 12];
rgb.y = pSrc[j + k + 13];
rgb.z = pSrc[j + k + 14];

y4 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;

rgb.x = pSrc[j + k + 15];
rgb.y = pSrc[j + k + 16];
rgb.z = pSrc[j + k + 17];

y5 = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;

pF.x = (v0 << 20) | (y0 << 10) | u0;
pF.y = (y2 << 20) | (u1 << 10) | y1;
pF.z = (u2 << 20) | (y3 << 10) | v1;
pF.w = (y5 << 20) | (v2 << 10) | y4;

k = tid * 8;
j *= 4;
j /= 9;
pDst[j + k + 0] = (uint32_t)(pF.x & 0x0000FFFF);
pDst[j + k + 1] = (uint32_t)(pF.x >> 16);
pDst[j + k + 2] = (uint32_t)(pF.y & 0x0000FFFF);
pDst[j + k + 3] = (uint32_t)(pF.y >> 16);
pDst[j + k + 4] = (uint32_t)(pF.z & 0x0000FFFF);
pDst[j + k + 5] = (uint32_t)(pF.z >> 16);
pDst[j + k + 6] = (uint32_t)(pF.w & 0x0000FFFF);
pDst[j + k + 7] = (uint32_t)(pF.w >> 16);
}
}