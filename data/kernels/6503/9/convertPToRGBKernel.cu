#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void convertPToRGBKernel(const uint16_t *dpSrc, uint8_t *tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uint32_t v0, y0, u0, y1;
uint16_t tt[6];
int nDstH = nDstHeight;
int nDstW = nSrcWidth / 2;
if (tid < nDstW && tidd < nDstH) {
int k = tid * 2;
int j = tidd * nSrcWidth;
y0 = (uint32_t)dpSrc[j + k + 0] * 1000;
y1 = (uint32_t)dpSrc[j + k + 1] * 1000;
k = tid;
j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth;
u0 = (uint32_t)dpSrc[j + k + 0];
j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth * 3 / 2;
v0 = (uint32_t)dpSrc[j + k + 0];

k = tid * 6;
j = tidd * nDstWidth * 3;
int r = 1407 * v0 - 720384, g = 716 * v0 + 345 * u0 - 543232, b = 1779 * u0 - 910848;
tt[0] = (y0 + r) / 1000;
tt[1] = (y0 - g) / 1000;
tt[2] = (y0 + b) / 1000;
tt[3] = (y1 + r) / 1000;
tt[4] = (y1 - g) / 1000;
tt[5] = (y1 + b) / 1000;

tt1[j + k + 0] = lookupTable[tt[0]];
tt1[j + k + 1] = lookupTable[tt[1]];
tt1[j + k + 2] = lookupTable[tt[2]];
tt1[j + k + 3] = lookupTable[tt[3]];
tt1[j + k + 4] = lookupTable[tt[4]];
tt1[j + k + 5] = lookupTable[tt[5]];
}
}