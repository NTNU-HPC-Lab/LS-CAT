#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void convertVToRGBKernel(const uint16_t *pV210, uint8_t *tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
uint16_t tt[6];
uint4 pF;
int nDstH = nDstHeight;
int nDstW = nSrcWidth / 8;

if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcWidth;
int k = tid * 8;
pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10) * 1000;
u0 = (uint32_t)(pF.x & 0x000003FF);
y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20) * 1000;
u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
y1 = (uint32_t)(pF.y & 0x000003FF) * 1000;
u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
y3 = (uint32_t)((pF.z & 0x000FFC00) >> 10) * 1000;
v1 = (uint32_t)(pF.z & 0x000003FF);
y5 = (uint32_t)((pF.w & 0x3FF00000) >> 20) * 1000;
v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
y4 = (uint32_t)(pF.w & 0x000003FF) * 1000;

k = tid * 18;
j *= 9;
j /= 4;
int r = 1407 * v0 - 720384, g = 716 * v0 + 345 * u0 - 543232, b = 1779 * u0 - 910848;
tt[0] = (y0 + r) / 1000;
tt[1] = (y0 - g) / 1000;
tt[2] = (y0 + b) / 1000;

tt[3] = (y1 + r) / 1000;
tt[4] = (y1 - g) / 1000;
tt[5] = (y1 + b) / 1000;

tt1[j + k + 0] = lookupTable[tt[0]];
tt1[j + k + 1] = lookupTable[tt[1]];
tt1[j + k + 2] = lookupTable[tt[2]];

tt1[j + k + 3] = lookupTable[tt[3]];
tt1[j + k + 4] = lookupTable[tt[4]];
tt1[j + k + 5] = lookupTable[tt[5]];

r = 1407 * v1 - 720384, g = 716 * v1 + 345 * u1 - 543232, b = 1779 * u1 - 910848;
tt[0] = (y2 + r) / 1000;
tt[1] = (y2 - g) / 1000;
tt[2] = (y2 + b) / 1000;

tt[3] = (y3 + r) / 1000;
tt[4] = (y3 - g) / 1000;
tt[5] = (y3 + b) / 1000;

tt1[j + k + 6] = lookupTable[tt[0]];
tt1[j + k + 7] = lookupTable[tt[1]];
tt1[j + k + 8] = lookupTable[tt[2]];

tt1[j + k + 9] = lookupTable[tt[3]];
tt1[j + k + 10] = lookupTable[tt[4]];
tt1[j + k + 11] = lookupTable[tt[5]];

r = 1407 * v2 - 720384, g = 716 * v2 + 345 * u2 - 543232, b = 1779 * u2 - 910848;
tt[0] = (y4 + r) / 1000;
tt[1] = (y4 - g) / 1000;
tt[2] = (y4 + b) / 1000;

tt[3] = (y5 + r) / 1000;
tt[4] = (y5 - g) / 1000;
tt[5] = (y5 + b) / 1000;

tt1[j + k + 12] = lookupTable[tt[0]];
tt1[j + k + 13] = lookupTable[tt[1]];
tt1[j + k + 14] = lookupTable[tt[2]];

tt1[j + k + 15] = lookupTable[tt[3]];
tt1[j + k + 16] = lookupTable[tt[4]];
tt1[j + k + 17] = lookupTable[tt[5]];
}
}