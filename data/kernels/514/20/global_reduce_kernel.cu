#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
int myId = threadIdx.x + blockDim.x * blockIdx.x;
int tid  = threadIdx.x;

// do reduction in global mem
for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
{
if (tid < s)
{
d_in[myId] += d_in[myId + s];
}
__syncthreads();        // make sure all adds at one stage are done!
}

// only thread 0 writes result for this block back to global mem
if (tid == 0)
{
d_out[blockIdx.x] = d_in[myId];
}
}