#include "hip/hip_runtime.h"
#include "includes.h"
/*
============================================================================
Name        : SpikeSorting.cu
Author      : John
Version     :
Copyright   :
Description : CUDA compute reciprocals
============================================================================
*/


static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* CUDA kernel that computes reciprocal values for a given vector
*/

/**
* Host function that copies the data and launches the work on GPU
*/
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
if (idx < vectorSize)
data[idx] = 1.0/data[idx];
}