#include "hip/hip_runtime.h"
#include "includes.h"

/*
waveform.cu:°üº¬µÄº¯ÊýÖ÷ÒªÊÇ¶ÔÓ¦SpikeDetect²¿·ÖµÄwaveformµÄÒ»Ð©²Ù×÷
º¯Êý×÷ÓÃÈçÏÂ£º
comps_wave()£º¶ÔÓÚdetect²¿·ÖÌáÈ¡µ½µÄcomponents£¬´Ó±ä»»ºóµÄ²¨ÐÎdata_tÖÐÌáÈ¡¶ÔÓ¦µÄwave
normalize()£º¶ÔÓÚ²¨ÐÎÖÐµÄµçÎ»Öµ£¬Í¨¹ý¸ßãÐÖµtsºÍµÍãÐÖµtw½øÐÐ¹éÒ»»¯£¬·½±ãÖ®ºó¼ÆËãmasksºÍ¼â·åµÄÖÐÐÄÊ±¼ä
compute_masks():¶ÔÓÚÃ¿Ò»¸öÌáÈ¡µ½µÄwave£¬¼ÆËãÆämasksµÄÖµ
*/
/*******************************************************copy the components to the wave**************************************************************/
/****************************************************normalize²Ù×÷*************************************************************/
/****************************************************compute_masks²Ù×÷*************************************************************/
__global__ void normalize(float *nor_ary, float *flit_ary,float tw,float ts, size_t N)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < N)
{
if (flit_ary[tid] >= ts) nor_ary[tid] = 1;
else if (nor_ary[tid] < tw) nor_ary[tid] = 0;
else nor_ary[tid] = (flit_ary[tid] - tw) / (ts - tw);
}
}