#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReplaceKernelA(const float* p_Input, float* p_Output, int p_Width, int p_Height, float hueRangeA, float hueRangeB, float hueRangeWithRollOffA, float hueRangeWithRollOffB, float satRangeA, float satRangeB, float satRolloff, float valRangeA, float valRangeB, float valRolloff, int OutputAlpha, int DisplayAlpha, float p_Black, float p_White) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
float hcoeff, scoeff, vcoeff;
float r, g, b, h, s, v;
r = p_Input[index];
g = p_Input[index + 1];
b = p_Input[index + 2];
float min = fmin(fmin(r, g), b);
float max = fmax(fmax(r, g), b);
v = max;
float delta = max - min;
if (max != 0.0f) {
s = delta / max;
} else {
s = 0.0f;
h = 0.0f;
}
if (delta == 0.0f) {
h = 0.0f;
} else if (r == max) {
h = (g - b) / delta;
} else if (g == max) {
h = 2 + (b - r) / delta;
} else {
h = 4 + (r - g) / delta;
}
h *= 1 / 6.0f;
if (h < 0.0f) {
h += 1.0f;
}
h *= 360.0f;
float h0 = hueRangeA;
float h1 = hueRangeB;
float h0mrolloff = hueRangeWithRollOffA;
float h1prolloff = hueRangeWithRollOffB;
if ( ( h1 < h0 && (h <= h1 || h0 <= h) ) || (h0 <= h && h <= h1) ) {
hcoeff = 1.0f;
} else {
float c0 = 0.0f;
float c1 = 0.0f;
if ( ( h0 < h0mrolloff && (h <= h0 || h0mrolloff <= h) ) || (h0mrolloff <= h && h <= h0) ) {
c0 = h0 == (h0mrolloff + 360.0f) || h0 == h0mrolloff ? 1.0f : !(( h0 < h0mrolloff && (h <= h0 || h0mrolloff <= h) ) || (h0mrolloff <= h && h <= h0)) ? 0.0f :
((h < h0mrolloff ? h + 360.0f : h) - h0mrolloff) / ((h0 < h0mrolloff ? h0 + 360.0f : h0) - h0mrolloff);
}
if ( ( h1prolloff < h1 && (h <= h1prolloff || h1 <= h) ) || (h1 <= h && h <= h1prolloff) ) {
c1 = !(( h1prolloff < h1 && (h <= h1prolloff || h1 <= h) ) || (h1 <= h && h <= h1prolloff)) ? 0.0f : h1prolloff == h1 ? 1.0f :
((h1prolloff < h1 ? h1prolloff + 360.0f : h1prolloff) - (h < h1 ? h + 360.0f : h)) / ((h1prolloff < h1 ? h1prolloff + 360.0f : h1prolloff) - h1);
}
hcoeff = fmax(c0, c1);
}
float s0 = satRangeA;
float s1 = satRangeB;
float s0mrolloff = s0 - satRolloff;
float s1prolloff = s1 + satRolloff;
if ( s0 <= s && s <= s1 ) {
scoeff = 1.0f;
} else if ( s0mrolloff <= s && s <= s0 ) {
scoeff = (s - s0mrolloff) / satRolloff;
} else if ( s1 <= s && s <= s1prolloff ) {
scoeff = (s1prolloff - s) / satRolloff;
} else {
scoeff = 0.0f;
}
float v0 = valRangeA;
float v1 = valRangeB;
float v0mrolloff = v0 - valRolloff;
float v1prolloff = v1 + valRolloff;
if ( (v0 <= v) && (v <= v1) ) {
vcoeff = 1.0f;
} else if ( v0mrolloff <= v && v <= v0 ) {
vcoeff = (v - v0mrolloff) / valRolloff;
} else if ( v1 <= v && v <= v1prolloff ) {
vcoeff = (v1prolloff - v) / valRolloff;
} else {
vcoeff = 0.0f;
}
float coeff = fmin(fmin(hcoeff, scoeff), vcoeff);
float A = OutputAlpha == 0 ? 1.0f : OutputAlpha == 1 ? hcoeff : OutputAlpha == 2 ? scoeff :
OutputAlpha == 3 ? vcoeff : OutputAlpha == 4 ? fmin(hcoeff, scoeff) : OutputAlpha == 5 ?
fmin(hcoeff, vcoeff) : OutputAlpha == 6 ? fmin(scoeff, vcoeff) : fmin(fmin(hcoeff, scoeff), vcoeff);
if (DisplayAlpha == 0)
A = coeff;
if (p_Black > 0.0f)
A = fmax(A - (p_Black * 4.0f) * (1.0f - A), 0.0f);
if (p_White > 0.0f)
A = fmin(A * (1.0f + p_White * 4.0f), 1.0f);
p_Output[index] = h;
p_Output[index + 1] = s;
p_Output[index + 2] = v;
p_Output[index + 3] = A;
}}