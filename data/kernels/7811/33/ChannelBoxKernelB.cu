#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChannelBoxKernelB(const float* p_Input, float* p_Output, int p_Width, int p_Height, int p_Display) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if ((x < p_Width) && (y < p_Height))
{
const int index = (y * p_Width + x) * 4;
p_Output[index] = p_Display == 1 ? p_Output[index + 3] : p_Output[index] * p_Output[index + 3] + p_Input[index] * (1.0f - p_Output[index + 3]);
p_Output[index + 1] = p_Display == 1 ? p_Output[index + 3] : p_Output[index + 1] * p_Output[index + 3] + p_Input[index + 1] * (1.0f - p_Output[index + 3]);
p_Output[index + 2] = p_Display == 1 ? p_Output[index + 3] : p_Output[index + 2] * p_Output[index + 3] + p_Input[index + 2] * (1.0f - p_Output[index + 3]);
p_Output[index + 3] = 1.0f;
}}