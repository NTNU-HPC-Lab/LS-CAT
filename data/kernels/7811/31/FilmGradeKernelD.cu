#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FilmGradeKernelD( float* p_Input, float* p_Output, int p_Width, int p_Height, float p_Pivot, int p_Display) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
float height = p_Height;
float width = p_Width;
float X = x;
float Y = y;
const float RES = width / 1920.0f;
float overlay = 0.0f;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
if (p_Display == 1) {
overlay = Y / height >= p_Pivot && Y / height <= p_Pivot + 0.005f * RES ? (fmodf(X, 2.0f) != 0.0f ? 1.0f : 0.0f) :
p_Output[index] >= (Y - 5.0f * RES) / height && p_Output[index] <= (Y + 5.0f * RES) / height ? 1.0f : 0.0f;
p_Output[index] = overlay;
}
if (p_Display == 2) {
overlay = Y / height >= p_Pivot && Y / height <= p_Pivot + 0.005f * RES ? (fmodf(X, 2.0f) != 0.0f ? 1.0f : 0.0f) :
p_Input[index] >= (Y - 5.0f * RES) / height && p_Input[index] <= (Y + 5.0f * RES) / height ? 1.0f : 0.0f;
p_Output[index] = overlay == 0.0f ? p_Output[index] : overlay;
}}}