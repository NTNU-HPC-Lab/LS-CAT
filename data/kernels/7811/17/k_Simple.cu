#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k_Simple( const float* p_Input, float* p_Output, int p_Width, int p_Height) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
p_Output[index] = p_Input[index];
p_Output[index + 1] = p_Input[index + 1];
p_Output[index + 2] = p_Input[index + 2];
p_Output[index + 3] = p_Input[index + 3];
}}