#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Prepare(float* p_Input, float* p_Output, int p_Width, int p_Height, int p_Display) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
float ramp = (float)x / (float)(p_Width - 1);
p_Output[index] = p_Display == 1 ? ramp : p_Input[index];
p_Output[index + 1] = p_Display == 1 ? ramp : p_Input[index + 1];
p_Output[index + 2] = p_Display == 1 ? ramp : p_Input[index + 2];
p_Output[index + 3] = 1.0f;
if (p_Display == 2) {
p_Input[index] = ramp;
p_Input[index + 1] = ramp;
p_Input[index + 2] = ramp;
}}}