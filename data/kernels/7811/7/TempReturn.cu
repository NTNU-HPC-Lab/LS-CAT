#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void Temp( float *R, float *G, float *B, float Temp) {
float r, g, b;
if (Temp <= 66.0f){
r = 255.0f;
} else {
r = Temp - 60.0f;
r = 329.698727446 * powf(r, -0.1332047592);
if(r < 0.0f){r = 0.0f;}
if(r > 255.0f){r = 255.0f;}
}
if (Temp <= 66.0f){
g = Temp;
g = 99.4708025861 * log(g) - 161.1195681661;
if(g < 0.0f){g = 0.0f;}
if(g > 255.0f){g = 255.0f;}
} else {
g = Temp - 60.0f;
g = 288.1221695283 * powf(g, -0.0755148492);
if(g < 0.0f){g = 0.0f;}
if(g > 255.0f){g = 255.0f;}
}
if(Temp >= 66.0f){
b = 255.0f;
} else {
if(Temp <= 19.0f){
b = 0.0f;
} else {
b = Temp - 10.0f;
b = 138.5177312231 * log(b) - 305.0447927307;
if(b < 0.0f){b = 0.0f;}
if(b > 255.0f){b = 255.0f;}
}
}
*R = r / 255.0f;
*G = g / 255.0f;
*B = b / 255.0f;
}
__global__ void TempReturn(float* p_Input, float* p_Temp, int p_Width, int p_Height)
{
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if ((x < p_Width) && (y < p_Height))
{
const int index = (y * p_Width + x) * 4;
p_Input[index + 2] = p_Temp[y * p_Width + x];
}}