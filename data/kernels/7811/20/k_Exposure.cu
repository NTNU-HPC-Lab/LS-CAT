#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k_Exposure( float* p_Input, int p_Width, int p_Height, float p_Exposure) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
p_Input[index] = p_Input[index] * exp2(p_Exposure);
p_Input[index + 1] = p_Input[index + 1] * exp2(p_Exposure);
p_Input[index + 2] = p_Input[index + 2] * exp2(p_Exposure);
}}