#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float Sat(float r, float g, float b){
float min = fmin(fmin(r, g), b);
float max = fmax(fmax(r, g), b);
float delta = max - min;
float S = max != 0.0f ? delta / max : 0.0f;
return S;
}
__global__ void FilmGradeKernelC( float* p_Input, int p_Width, int p_Height, float p_ContR, float p_ContG, float p_ContB, float p_SatR, float p_SatG, float p_SatB, float p_ContP) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
float contR = (p_Input[index] - p_ContP) * p_ContR + p_ContP;
float contG = (p_Input[index + 1] - p_ContP) * p_ContG + p_ContP;
float contB = (p_Input[index + 2] - p_ContP) * p_ContB + p_ContP;
float luma = contR * 0.2126f + contG * 0.7152f + contB * 0.0722f;
float outR = (1.0f - (p_SatR * 0.2126f + p_SatG * 0.7152f + p_SatB * 0.0722f)) * luma + contR * p_SatR;
float outG = (1.0f - (p_SatR * 0.2126f + p_SatG * 0.7152f + p_SatB * 0.0722f)) * luma + contG * p_SatG;
float outB = (1.0f - (p_SatR * 0.2126f + p_SatG * 0.7152f + p_SatB * 0.0722f)) * luma + contB * p_SatB;
p_Input[index] = outR;
p_Input[index + 1] = outG;
p_Input[index + 2] = outB;
}}