#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FilmGradeKernelA( float* p_Input, int p_Width, int p_Height, float p_Exp) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if(x < p_Width && y < p_Height) {
const int index = (y * p_Width + x) * 4;
p_Input[index] = p_Input[index] + p_Exp * 0.01f;
}}