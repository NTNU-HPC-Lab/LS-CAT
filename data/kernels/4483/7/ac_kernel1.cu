#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ac_kernel1 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {

//int idx = blockIdx.x * blockDim.x + threadIdx.x;
int effective_pitch = pitch / sizeof ( int );

int charactersPerBlock = n / numBlocks;

int startBlock = blockIdx.x * charactersPerBlock;
int stopBlock = startBlock + charactersPerBlock;

int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;

int startThread = startBlock + charactersPerThread * threadIdx.x;
int stopThread;
if( blockIdx.x == numBlocks -1 && threadIdx.x==blockDim.x-1)
stopThread = n - 1;
else stopThread = startThread + charactersPerThread + m-1;

int r = 0, s;

int column;

//cuPrintf("Working from %i to %i chars %i\n", startThread, stopThread, charactersPerThread);

for ( column = startThread; ( column < stopThread && column < n ); column++ ) {

while ( ( s = d_state_transition[r * effective_pitch + (d_text[column]-(unsigned char)'A')] ) == -1 )
r = d_state_supply[r];
r = s;

d_out[column] = d_state_final[r];
}
}