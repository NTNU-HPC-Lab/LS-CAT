#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "global_memory_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *d_go_to_state = NULL;
hipMalloc(&d_go_to_state, XSIZE*YSIZE);
unsigned int *d_failure_state = NULL;
hipMalloc(&d_failure_state, XSIZE*YSIZE);
unsigned int *d_output_state = NULL;
hipMalloc(&d_output_state, XSIZE*YSIZE);
unsigned char *d_text = NULL;
hipMalloc(&d_text, XSIZE*YSIZE);
unsigned int *d_out = NULL;
hipMalloc(&d_out, XSIZE*YSIZE);
size_t pitch = 2;
int m = 2;
int n = XSIZE*YSIZE;
int p_size = XSIZE*YSIZE;
int alphabet = 2;
int num_blocks = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
global_memory_kernel<<<gridBlock,threadBlock>>>(d_go_to_state,d_failure_state,d_output_state,d_text,d_out,pitch,m,n,p_size,alphabet,num_blocks);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
global_memory_kernel<<<gridBlock,threadBlock>>>(d_go_to_state,d_failure_state,d_output_state,d_text,d_out,pitch,m,n,p_size,alphabet,num_blocks);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
global_memory_kernel<<<gridBlock,threadBlock>>>(d_go_to_state,d_failure_state,d_output_state,d_text,d_out,pitch,m,n,p_size,alphabet,num_blocks);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}