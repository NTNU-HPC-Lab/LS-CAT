#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_memory_kernel(int *d_go_to_state, unsigned int *d_failure_state, unsigned int *d_output_state, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int num_blocks ) {

int idx = blockIdx.x * blockDim.x + threadIdx.x;
int effective_pitch = pitch / sizeof ( int );

int chars_per_block = n / num_blocks;

int start_block = blockIdx.x * chars_per_block;
int stop_block = start_block + chars_per_block;

int chars_per_thread = ( stop_block - start_block ) / blockDim.x;

int start_thread = start_block + chars_per_thread * threadIdx.x;
int stop_thread;
if( blockIdx.x == num_blocks -1 && threadIdx.x==blockDim.x-1)
stop_thread = n - 1;
else stop_thread = start_thread + chars_per_thread + m-1;

int r = 0, s;

int column;

for ( column = start_thread; ( column < stop_thread && column < n ); column++ ) {

while ( ( s = d_go_to_state[r * effective_pitch + (d_text[column]-(unsigned char)'A')] ) == -1 )
r = d_failure_state[r];
r = s;

d_out[idx] += d_output_state[r];
}
}