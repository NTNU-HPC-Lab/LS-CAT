#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Cuda kernels that does the heavy work
*/




////////////////////////////////////////////////////////////////////////////////
// declaration, forward
extern "C" void check_cuda(hipError_t result, char const *const func, const char *const file, int const line);


__global__ void render_init_kernel(int max_x, int max_y, hiprandState *rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if ((i >= max_x) || (j >= max_y)) return;
int pixel_index = j * max_x + i;
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}