#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Counting(int* HalfData, int HalfDataSize, int N)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i<HalfDataSize)
{
HalfData[i] *= N;
}
}