#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif
float a[1024][1024], b[1024][1024], c[1024][1024];




// Now launch your kernel using the appropriate macro:

// Now launch your kernel using the appropriate macro:
//kernel KERNEL_ARGS2(dim3(nBlockCount), dim3(nThreadCount)) (param1);



//matrix multiplication on GPU
__global__ void MMul(float*m, float*d, float*p, int n) {
int r = blockIdx.y*blockDim.y + threadIdx.y;// row
int c = blockIdx.x*blockDim.x + threadIdx.x;//column
float p_sum = 0;

for (int i = 0; i < n; i++) {
p_sum = +m[r*n + i] * d[i*n + c];
}
p[r*n + c] = p_sum;
}