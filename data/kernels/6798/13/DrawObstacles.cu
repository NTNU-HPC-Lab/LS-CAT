#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawObstacles(uchar4 *ptr, int* indices, int size) {

int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

while (thread_id < size) {
int index = indices[thread_id];
ptr[index].x = 0;
ptr[index].y = 0;
ptr[index].z = 0;
ptr[index].w = 255;

thread_id += blockDim.x * gridDim.x;
}
}