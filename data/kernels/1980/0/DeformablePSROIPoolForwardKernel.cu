#include "hip/hip_runtime.h"
#include "includes.h"
/*!
* Copyright (c) 2017 Microsoft
* Licensed under The MIT License [see LICENSE for details]
* \file deformable_psroi_pooling.cu
* \brief
* \author Yi Li, Guodong Zhang, Jifeng Dai
*/
/***************** Adapted by Charles Shang *********************/

#define CUDA_KERNEL_LOOP(i, n)                        \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
i < (n);                                       \
i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
__device__ float bilinear_interp( const float *data, const float x, const float y, const int width, const int height)
{
int x1 = floor(x);
int x2 = ceil(x);
int y1 = floor(y);
int y2 = ceil(y);
float dist_x = (float)(x - x1);
float dist_y = (float)(y - y1);
float value11 = data[y1 * width + x1];
float value12 = data[y2 * width + x1];
float value21 = data[y1 * width + x2];
float value22 = data[y2 * width + x2];
float value = (1 - dist_x) * (1 - dist_y) * value11 + (1 - dist_x) * dist_y * value12 + dist_x * (1 - dist_y) * value21 + dist_x * dist_y * value22;
return value;
}
__device__ double bilinear_interp( const double *data, const double x, const double y, const int width, const int height)
{
int x1 = floor(x);
int x2 = ceil(x);
int y1 = floor(y);
int y2 = ceil(y);
double dist_x = (double)(x - x1);
double dist_y = (double)(y - y1);
double value11 = data[y1 * width + x1];
double value12 = data[y2 * width + x1];
double value21 = data[y1 * width + x2];
double value22 = data[y2 * width + x2];
double value = (1 - dist_x) * (1 - dist_y) * value11 + (1 - dist_x) * dist_y * value12 + dist_x * (1 - dist_y) * value21 + dist_x * dist_y * value22;
return value;
}
__global__ void DeformablePSROIPoolForwardKernel( const int count, const double *bottom_data, const double spatial_scale, const int channels, const int height, const int width, const int pooled_height, const int pooled_width, const double *bottom_rois, const double *bottom_trans, const int no_trans, const double trans_std, const int sample_per_part, const int output_dim, const int group_size, const int part_size, const int num_classes, const int channels_each_class, double *top_data, double *top_count)
{
CUDA_KERNEL_LOOP(index, count)
{
// The output is in order (n, ctop, ph, pw)
int pw = index % pooled_width;
int ph = (index / pooled_width) % pooled_height;
int ctop = (index / pooled_width / pooled_height) % output_dim;
int n = index / pooled_width / pooled_height / output_dim;

// [start, end) interval for spatial sampling
const double *offset_bottom_rois = bottom_rois + n * 5;
int roi_batch_ind = offset_bottom_rois[0];
double roi_start_w = (double)(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
double roi_start_h = (double)(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
double roi_end_w = (double)(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
double roi_end_h = (double)(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

// Force too small ROIs to be 1x1
double roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
double roi_height = max(roi_end_h - roi_start_h, 0.1);

// Compute w and h at bottom
double bin_size_h = roi_height / (double)(pooled_height);
double bin_size_w = roi_width / (double)(pooled_width);

double sub_bin_size_h = bin_size_h / (double)(sample_per_part);
double sub_bin_size_w = bin_size_w / (double)(sample_per_part);

int part_h = floor((double)(ph) / pooled_height * part_size);
int part_w = floor((double)(pw) / pooled_width * part_size);
int class_id = ctop / channels_each_class;
double trans_x = no_trans ? (double)(0) : bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w] * trans_std;
double trans_y = no_trans ? (double)(0) : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * trans_std;

double wstart = (double)(pw)*bin_size_w + roi_start_w;
wstart += trans_x * roi_width;
double hstart = (double)(ph)*bin_size_h + roi_start_h;
hstart += trans_y * roi_height;

double sum = 0;
int count = 0;
int gw = floor((double)(pw)*group_size / pooled_width);
int gh = floor((double)(ph)*group_size / pooled_height);
gw = min(max(gw, 0), group_size - 1);
gh = min(max(gh, 0), group_size - 1);

const double *offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;
for (int ih = 0; ih < sample_per_part; ih++)
{
for (int iw = 0; iw < sample_per_part; iw++)
{
double w = wstart + iw * sub_bin_size_w;
double h = hstart + ih * sub_bin_size_h;
// bilinear interpolation
if (w < -0.5 || w > width - 0.5 || h < -0.5 || h > height - 0.5)
{
continue;
}
w = min(max(w, 0.), width - 1.);
h = min(max(h, 0.), height - 1.);
int c = (ctop * group_size + gh) * group_size + gw;
double val = bilinear_interp(offset_bottom_data + c * height * width, w, h, width, height);
sum += val;
count++;
}
}
top_data[index] = count == 0 ? (double)(0) : sum / count;
top_count[index] = count;
}
}