#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int order, const int matrices, double * C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

for (int b=0; b<matrices; ++b) {
if ((i<order) && (j<order)) {
C[b*order*order+i*order+j] = 0;
}
}
}