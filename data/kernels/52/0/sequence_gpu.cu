#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sequence_gpu(int *d_ptr, int length)
{
int elemID = blockIdx.x * blockDim.x + threadIdx.x;

if (elemID < length)
{
d_ptr[elemID] = elemID;
}
}