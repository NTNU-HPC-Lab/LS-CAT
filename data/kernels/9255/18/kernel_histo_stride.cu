#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_stride( unsigned int *ct, unsigned int *histo){


int i = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while( i < constant_n_hits*constant_n_test_vertices ){
atomicAdd( &histo[ct[i]], 1);
i += stride;
}


}