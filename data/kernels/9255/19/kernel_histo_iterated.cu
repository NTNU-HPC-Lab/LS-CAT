#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_iterated( unsigned int *ct, unsigned int *histo, unsigned int offset ){


extern __shared__ unsigned int temp[];
unsigned int index = threadIdx.x + offset;
temp[index] = 0;
__syncthreads();
int i = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int size = blockDim.x * gridDim.x;
unsigned int max = constant_n_hits*constant_n_test_vertices;
while( i < max ){
atomicAdd( &temp[ct[i]], 1);
i += size;
}
__syncthreads();
atomicAdd( &(histo[index]), temp[index] );


}