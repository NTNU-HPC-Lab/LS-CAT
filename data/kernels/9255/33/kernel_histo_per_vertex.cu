#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_per_vertex( unsigned int *ct, unsigned int *histo){

// get unique id for each thread in each block
unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
unsigned int tid_y = threadIdx.y + blockDim.y*blockIdx.y;

if( tid_x >= constant_n_test_vertices ) return;

unsigned int vertex_offset = tid_x*constant_n_hits;
unsigned int bin;
unsigned int stride = blockDim.y*gridDim.y;
unsigned int ihit = vertex_offset + tid_y;

while( ihit<vertex_offset+constant_n_hits){

bin = ct[ihit];
//histo[bin]++;
atomicAdd( &histo[bin], 1);
ihit += stride;

}
__syncthreads();
}