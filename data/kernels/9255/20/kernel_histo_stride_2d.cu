#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_stride_2d( unsigned int *ct, unsigned int *histo){

// get unique id for each thread in each block
unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
unsigned int tid_y = threadIdx.y + blockDim.y*blockIdx.y;

unsigned int size = blockDim.x * gridDim.x;
unsigned int max = constant_n_hits*constant_n_test_vertices;

// map the two 2D indices to a single linear, 1D index
int tid = tid_y * size + tid_x;

/*
unsigned int vertex_index = (int)(tid/constant_n_time_bins);
unsigned int time_index = tid - vertex_index * constant_n_time_bins;

// skip if thread is assigned to nonexistent vertex
if( vertex_index >= constant_n_test_vertices ) return;

// skip if thread is assigned to nonexistent hit
if( time_index >= constant_n_time_bins ) return;

unsigned int vertex_block = constant_n_time_bins*vertex_index;

unsigned int vertex_block2 = constant_n_PMTs*vertex_index;
*/

unsigned int stride = blockDim.y * gridDim.y*size;

while( tid < max ){
atomicAdd( &histo[ct[tid]], 1);
tid += stride;
}


}