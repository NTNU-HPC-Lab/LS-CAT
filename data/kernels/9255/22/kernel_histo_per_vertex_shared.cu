#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_per_vertex_shared( unsigned int *ct, unsigned int *histo){
// get unique id for each thread in each block
unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
unsigned int tid_y = threadIdx.y + blockDim.y*blockIdx.y;

if( tid_x >= constant_n_test_vertices ) return;

unsigned int vertex_offset = tid_x*constant_n_hits;
unsigned int bin;
unsigned int stride = blockDim.y*gridDim.y;
unsigned int stride_block = blockDim.y;
unsigned int ihit = vertex_offset + tid_y;
unsigned int time_offset = tid_x*constant_n_time_bins;

unsigned int local_ihit = threadIdx.y;
extern __shared__ unsigned int temp[];
while( local_ihit<constant_n_time_bins ){
temp[local_ihit] = 0;
local_ihit += stride_block;
}

__syncthreads();

while( ihit<vertex_offset+constant_n_hits){

bin = ct[ihit];
atomicAdd(&temp[bin - time_offset],1);
ihit += stride;

}

__syncthreads();

local_ihit = threadIdx.y;
while( local_ihit<constant_n_time_bins ){
atomicAdd( &histo[local_ihit+time_offset], temp[local_ihit]);
local_ihit += stride_block;
}


}