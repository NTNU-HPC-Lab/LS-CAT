#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_one_thread_one_vertex( unsigned int *ct, unsigned int *histo ){


// get unique id for each thread in each block
unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;

unsigned int vertex_index = tid_x;
unsigned int bin ;
unsigned int max = constant_n_test_vertices*constant_n_hits;
unsigned int size = vertex_index * constant_n_hits;

for( unsigned int ihit=0; ihit<constant_n_hits; ihit++){
bin = size + ihit;
if( bin < max)
atomicAdd(&histo[ct[bin]],1);
}

}