#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inc (int n, float* a) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) {
a[i] += 1;
}
};