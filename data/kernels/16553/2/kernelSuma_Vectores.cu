#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelSuma_Vectores(float* array_A, float* array_B, int _size){
int idx= blockIdx.x*blockDim.x+threadIdx.x;
if(idx<_size){
array_A[idx] = array_A[idx] + array_B[idx];
}
}