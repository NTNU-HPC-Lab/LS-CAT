#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelVector_x_constant( float* arr, int n, int k )
{
//Obtengo el indice del hilo fisico
int idx = blockIdx.x * blockDim.x + threadIdx.x;

//Mientras el hilo sea valido para la operaci�n
if( idx<n )
{
//Multiplico el elemento por la constante
arr[ idx ] = arr[ idx ] * k;
}
}