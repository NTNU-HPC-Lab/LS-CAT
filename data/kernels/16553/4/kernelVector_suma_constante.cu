#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelVector_suma_constante(float* array, int _size, int _constant){
int idx= blockIdx.x * blockDim.x + threadIdx.x;
if(idx < _size){
array[idx] = array[idx]+_constant;
}
}