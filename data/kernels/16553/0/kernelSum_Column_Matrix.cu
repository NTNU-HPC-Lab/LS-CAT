#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
const int DIMBLOCKX=32;
//DEVICE



//HOST
__global__ void kernelSum_Column_Matrix(float* matrix, float* array, int tam){
__shared__ float shareMatrix[DIMBLOCKX];

float value=0;
int col=blockIdx.x;
int step= tam/blockDim.x;
int posIni= col*tam+threadIdx.x*step;
for(int i=0;i<step;i++){
value=value+matrix[posIni+i];
}

shareMatrix[threadIdx.x]=value;
__syncthreads();

if(threadIdx.x==0){
for(int j=1;j<blockDim.x;j++){
shareMatrix[0]=shareMatrix[0]+shareMatrix[j];
}
array[blockIdx.x]=shareMatrix[0];
}
}