#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd1(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
__shared__ int uni;
if (threadIdx.x == 0)
uni = uniforms[blockIdx.x + blockOffset];

unsigned int address = __mul24(blockIdx.x, blockDim.x) + baseIndex + threadIdx.x;

__syncthreads();

// note one add per thread
g_data[address] += uni;
}