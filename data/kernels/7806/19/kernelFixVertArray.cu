#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFixVertArray(int *ctriangles, int nTris, int *cvertarr)

{
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
if (x >= nTris)
return ;

int v0 = ctriangles[x * 9 + 4];
int v1 = ctriangles[x * 9 + 5];
int v2 = ctriangles[x * 9 + 3];

ctriangles[x * 9 + 6] = atomicExch(&cvertarr[v0], (x << 2));
ctriangles[x * 9 + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1);
ctriangles[x * 9 + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2);
}