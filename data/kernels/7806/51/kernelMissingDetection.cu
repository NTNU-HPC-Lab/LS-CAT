#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMissingDetection(int nVerts, int *cactive, int *cvertarr) {
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

// Check for missing sites
if (x < nVerts && cvertarr[x] < 0)
cactive[x] = 0;
}