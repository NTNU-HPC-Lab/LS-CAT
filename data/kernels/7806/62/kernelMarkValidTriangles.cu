#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMarkValidTriangles(short *cnewtri, int *cvalid, int nTris)
{
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

if (x >= nTris)
return ;

cvalid[x] = (cnewtri[x] >= 0) ? 1 : 0;
}