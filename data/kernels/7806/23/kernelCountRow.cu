#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCountRow(int *voronoiPtr, short2 *patternPtr, int *count, int width, int min, int max, int *cboundary) {
// Get the row we are working on
int x = blockIdx.x * blockDim.x + threadIdx.x;

// Collect the boundary (up, left, down, right)
if (x > 0 && x <= max) {
cboundary[width * 0 + x] = voronoiPtr[min * width + x];
cboundary[width * 1 + x] = voronoiPtr[x * width + min];
cboundary[width * 2 + x] = voronoiPtr[max * width + x];
cboundary[width * 3 + x] = voronoiPtr[x * width + max];
}

// Actual counting
if (x < min || x >= max)
return ;

int xwidth = x * width;
int result = 0;
short2 t = patternPtr[xwidth + min];

// Keep jumping and counting
while (t.y > 0 && t.y < max) {
result += 1 + (t.x >> 2);
t = patternPtr[xwidth + t.y + 1];
}

count[x] = result;
}