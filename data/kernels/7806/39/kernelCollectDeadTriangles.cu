#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCollectDeadTriangles(int *cdeadTri, short *cnewtri, int *cmarker, int nTris) {
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

if (x >= nTris || cnewtri[x] >= 0)
return ;

int id = cmarker[x];

cdeadTri[id] = x;
}