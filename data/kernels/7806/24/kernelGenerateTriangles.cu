#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGenerateTriangles(int *voronoiPtr, short2 *patternPtr, int3 *ctriangles, int *offset, int width, int min, int max) {
int x = blockIdx.x * blockDim.x + threadIdx.x;

if (x < min || x >= max)
return ;

int xwidth = x * width;
short2 pattern = patternPtr[xwidth + min];
int i0, i1, i2, i3;
int3 *pT = &ctriangles[offset[x-1]];

// Jump through all voronoi vertices in a texture row
while (pattern.y > 0 && pattern.y < max) {
i0 = voronoiPtr[xwidth + pattern.y];
i1 = voronoiPtr[xwidth + pattern.y + 1];
i2 = voronoiPtr[xwidth + width + pattern.y + 1];
i3 = voronoiPtr[xwidth + width + pattern.y];

if (pattern.x == 0) *pT = make_int3(i3, i1, i2);
if (pattern.x == 1) *pT = make_int3(i0, i2, i3);
if (pattern.x == 2) *pT = make_int3(i1, i3, i0);
if (pattern.x == 3) *pT = make_int3(i2, i0, i1);
if (pattern.x == 4) {
// Generate 2 triangles.
// Since the hole is convex, no need to do CCW test
*pT = make_int3(i2, i0, i1); pT++;
*pT = make_int3(i3, i0, i2);
}

pattern = patternPtr[xwidth + pattern.y + 1];
pT++;
}
}