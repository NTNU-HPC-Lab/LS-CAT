#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMarkDeadTriangles(int *cmarker, short *cnewtri, int nTris) {
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

if (x >= nTris)
return ;

cmarker[x] = (cnewtri[x] >= 0 ? 0 : 1);
}