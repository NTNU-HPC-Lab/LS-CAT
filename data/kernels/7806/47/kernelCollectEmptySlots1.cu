#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCollectEmptySlots1(short *cnewtri, int *cprefix, int *cempty, int nTris)
{
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

if (x >= nTris || cnewtri[x] >= 0)
return ;

int id = x - cprefix[x];

cempty[id] = x;
}