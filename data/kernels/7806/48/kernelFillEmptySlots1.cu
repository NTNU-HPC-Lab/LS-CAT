#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFillEmptySlots1(short *cnewtri, int *cprefix, int *cempty, int *ctriangles, int nTris, int newnTris, int offset)
{
int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

if (x >= nTris || cnewtri[x] < 0)
return ;

int value;

if (x < newnTris)
value = x;
else {
value = cempty[cprefix[x] - offset];

for (int i = 0; i < 9; i++)
ctriangles[value * 9 + i] = ctriangles[x * 9 + i];
}

cprefix[x] = value;
}