#include "hip/hip_runtime.h"
#include "includes.h"

//#define ITEM_COUNT 2
#define _PI 3.14159265358979323846
#define _PI2 1.57079632679489661923
#define _RAD 6372795





using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void geo_invert(double2* d_dot1, double2* d_dot2, double* d_dist, double* d_azimut, long count)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < count)
{
d_dot1[idx].x = d_dot1[idx].x * _PI / 180;	//lat1
d_dot1[idx].y = d_dot1[idx].y * _PI / 180;	//lng1
d_dot2[idx].x = d_dot2[idx].x * _PI / 180;	//lat2
d_dot2[idx].y = d_dot2[idx].y * _PI / 180;	//lng2

double cl1, cl2, sl1, sl2, delta, cdelta, sdelta;
cl1 = cos(d_dot1[idx].x);
cl2 = cos(d_dot2[idx].x);
sl1 = sin(d_dot1[idx].x);
sl2 = sin(d_dot2[idx].x);
delta = d_dot2[idx].y - d_dot1[idx].y;
cdelta = cos(delta);
sdelta = sin(delta);

double x, y, z, ad, z2;
y = sqrt(pow(cl2*sdelta, 2) + pow(cl1*sl2 - sl1*cl2*cdelta, 2));
x = sl1*sl2 + cl1*cl2*cdelta;
ad = atan(y / x);
d_dist[idx] = ad * _RAD;

x = (cl1*sl2) - (sl1*cl2*cdelta);
y = sdelta*cl2;

if (x == 0)
{
if (y > 0)
z = -90;
else if (y < 0)
z = 90;
else if (y == 0)
z = 0;
}
else
{
z = atan(-y / x) * 180 / _PI;
if (x < 0)
{
z = z + 180;
}
}

z2 = z + 180.0f;

while (z2 >= 360)
{
z2 = z2 - 360;
}

z2 = z2 - 180;


z2 = -z2 * _PI / 180;
double anglerad2;
anglerad2 = z2 - ((2 * _PI) * floor(z2 / (2 * _PI)));
d_azimut[idx] = anglerad2 * 180 / _PI;


}
}