#include "hip/hip_runtime.h"
#include "includes.h"

//#define ITEM_COUNT 2
#define _PI 3.14159265358979323846
#define _PI2 1.57079632679489661923
#define _RAD 6372795





using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void d_cudainit(int *a, int *b)
{
int i = threadIdx.x;
if (i==1)
{
b[i] = a[i] * 2;
}
}