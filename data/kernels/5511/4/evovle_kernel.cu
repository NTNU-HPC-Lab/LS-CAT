#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
// Achieve indexng on 2D blocks
int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
// Thread calculates its global id
int id = ix * (N+2) + iy;

int neighbors;

if (ix <= N && iy <= N) {
neighbors = oldGen[id+(N+2)] + oldGen[id-(N+2)]     //lower upper
+ oldGen[id+1] + oldGen[id-1]           //right left
+ oldGen[id+(N+3)] + oldGen[id-(N+3)]   //diagonals
+ oldGen[id-(N+1)] + oldGen[id+(N+1)];

char cell  = oldGen[id];
newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Fill in the cells

// Terminating Checkings
if (newGen[id] != 0) (*allzeros)++;             // Check if all cells are dead
if (newGen[id] != oldGen[id]) (*change)++;      // Check if life stayed the same

}
}