#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
// Global
int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;       //Different indexing as we declared more blocks (see SideGrid)
int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
int id = ix * (N+2) + iy;

int i = threadIdx.x;
int j = threadIdx.y;
int neighbors;

// Declare the shared memory on a per block level
__shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];

// Copy cells into shared memory
if (ix <= N+1 && iy <= N+1)
oldGen_shared[i][j] = oldGen[id];           //Copy each cell and in the sides of shared array the blocks' neighbors

// Sync threads on block
__syncthreads();

if (ix <= N && iy <= N) {
if(i != 0 && i != (blockDim.y-1) && j != 0 && j != (blockDim.x-1)) {

// Get the number of neighbors for a given oldGen point
neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j]         //lower upper
+ oldGen_shared[i][j+1] + oldGen_shared[i][j-1]           //right left
+ oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1]       //diagonals
+ oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];

char cell  = oldGen_shared[i][j];
newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Fill in  the cells

// Terminating Checkings
if (newGen[id] != 0) (*allzeros)++;        // Check if all cells are dead
if (newGen[id] != oldGen[id]) (*change)++; // Check if life stayed the same
}
}
}