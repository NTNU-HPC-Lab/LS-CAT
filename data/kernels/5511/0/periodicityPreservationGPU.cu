#include "hip/hip_runtime.h"
#include "includes.h"

#define BUFSIZE 64
#define BLOCK_SIZE 16

// Perdiodicty Preservation retains our periodicity
// Runs on CPU
__global__ void periodicityPreservationGPU(int N, char *cells)
{
int i;
//rows
for (i = 1; i <= N; ++i)
{
//Copy first real row to bottom extra row
cells[(N+2)*(N+1)+i] = cells[(N+2)+i];
//Copy last real row to top extra row
cells[i] = cells[(N+2)*N + i];
}
//cols
for (i = 0; i <= N+1; ++i)
{
//Copy first real column to right last extra column
cells[i*(N+2)+N+1] = cells[i*(N+2)+1];
//Copy last real column to left last extra column
cells[i*(N+2)] = cells[i*(N+2) + N];
}
}