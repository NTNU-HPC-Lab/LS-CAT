#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

// ==========
// Macro taken from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__device__ int getElementsPerUnit(int total, int number_of_units) {
int elements_per_unit = total / number_of_units;
double remains = total % number_of_units;

if(remains != 0) {
elements_per_unit += 1;
}

return elements_per_unit;
}
__global__ void normKernel(double *vectors, int size, double *results, int vector_size) {
int vectors_per_block = getElementsPerUnit(size, gridDim.x);

// Get range of vectors we will work with
int start = blockIdx.x * vectors_per_block;
int end = start + vectors_per_block;

if(end > size) {
end = size;
}

for(int vec_index = start; vec_index < end; vec_index++) {
for(int i = 0; i < vector_size; i++) {
results[vec_index] += pow(vectors[vec_index*vector_size + i], 2);
}

results[vec_index] = sqrt(results[vec_index]);
}
}