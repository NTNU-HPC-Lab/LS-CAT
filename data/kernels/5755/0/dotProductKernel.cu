#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

// ==========
// Macro taken from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__device__ double dotProduct(double *a, double *b, int size) {
double result = 0;

for(int i = 0; i < size; i++) {
result += a[i] * b[i];
}

return result;
}
__device__ int getElementsPerUnit(int total, int number_of_units) {
int elements_per_unit = total / number_of_units;
double remains = total % number_of_units;

if(remains != 0) {
elements_per_unit += 1;
}

return elements_per_unit;
}
__global__ void dotProductKernel(double *a, double *b, int a_size, int b_size, double *results, int vector_size) {
int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

// Get range of 'a' vectors we will work with
int a_start = blockIdx.x * a_vectors_per_block;
int a_end = a_start + a_vectors_per_block;

if(a_end > a_size) {
a_end = a_size;
}

// Get range of 'b' vectors we will work with
int b_start = threadIdx.x * b_vectors_per_thread;
int b_end = b_start + b_vectors_per_thread;

if(b_end > b_size) {
b_end = b_size;
}

if(a_start < a_size && b_start < b_size) {
for(int a_index = a_start; a_index < a_end; a_index++) {
for(int b_index = b_start; b_index < b_end; b_index++) {
results[a_index*b_size + b_index] = dotProduct(&a[a_index*vector_size], &b[b_index*vector_size], vector_size);
}
}
}
}