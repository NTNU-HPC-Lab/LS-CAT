#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

// ==========
// Macro taken from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__device__ int getElementsPerUnit(int total, int number_of_units) {
int elements_per_unit = total / number_of_units;
double remains = total % number_of_units;

if(remains != 0) {
elements_per_unit += 1;
}

return elements_per_unit;
}
__global__ void cosineSimilarityKernel(double *dot_products, int a_size, int b_size, double *a_norms, double *b_norms, double *results) {
int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

int a_start = blockIdx.x * a_vectors_per_block;
int a_end = a_start + a_vectors_per_block;

if(a_end > a_size) {
a_end = a_size;
}

int b_start = threadIdx.x * b_vectors_per_thread;
int b_end = b_start + b_vectors_per_thread;

if(b_end > b_size) {
b_end = b_size;
}

for(int a_index = a_start; a_index < a_end; a_index++) {
for(int b_index = b_start; b_index < b_end; b_index++) {
results[a_index*b_size + b_index] = (double) dot_products[a_index*b_size + b_index] / (a_norms[a_index] * b_norms[b_index]);
}
}
}