#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Interleaved_Addressing_Shared(float* data, int data_size){
int idx = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ float sdata[MAX_CUDA_THREADS_PER_BLOCK];
if (idx < data_size){

/*copy to shared memory*/
sdata[threadIdx.x] = data[idx];
__syncthreads();

for(int stride=1; stride < blockDim.x; stride *= 2) {
if (threadIdx.x % (2*stride) == 0) {
float lhs = sdata[threadIdx.x];
float rhs = sdata[threadIdx.x + stride];
sdata[threadIdx.x] = lhs < rhs ? rhs : lhs;
}
__syncthreads();
}
}
if (idx == 0) data[0] = sdata[0];
}