#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Interleaved_Addressing_Global(float* data, int data_size){
int idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < data_size){
for(int stride=1; stride < data_size; stride *= 2) {
if (idx % (2*stride) == 0) {
float lhs = data[idx];
float rhs = data[idx + stride];
data[idx] = lhs < rhs ? rhs : lhs;
}
__syncthreads();
}
}
}