#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Sequential_Addressing_Shared(float* data, int data_size){
int idx = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ float sdata[MAX_CUDA_THREADS_PER_BLOCK];
if (idx < data_size){

/*copy to shared memory*/
sdata[threadIdx.x] = data[idx];
__syncthreads();

for(int stride=blockDim.x/2; stride > 0; stride /= 2) {
if (threadIdx.x < stride) {
float lhs = sdata[threadIdx.x];
float rhs = sdata[threadIdx.x + stride];
sdata[threadIdx.x] = lhs < rhs ? rhs : lhs;
}
__syncthreads();
}
}
if (idx == 0) data[0] = sdata[0];
}