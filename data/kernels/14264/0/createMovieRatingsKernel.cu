#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createMovieRatingsKernel(const float *weights, const float *initial_hidden_feature_probs, float* movie_rating_probs, int num_movies, int num_hidden_features) {

// weights[NUM_MOVIES][5][NUM_FEATURES]
// initial_hidden_feature_probs[NUM_FEATURES]
// final_movie_ratings[NUM_MOVIES][5]
//
// movie_rating_index = movie_id * 5 + rating_id
//      (index of current movie_id/rating_id pair)
unsigned int movie_rating_id = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int hidden_id = 0;
float dot_prod; // Temporary, local dot product variable
while (movie_rating_id < num_movies * 5) {
dot_prod = 0.00; // Initialize the dot product to 0

for (hidden_id = 0; hidden_id < num_hidden_features; hidden_id++) {
// Indexing: weights[movie_id][rating][feature_id]
// movie_id - [1, 17771]
// rating - [0, 4]
// feature_id - [0, 99]
// Do the dot product
dot_prod += weights[movie_rating_id*num_hidden_features
+ hidden_id]
* initial_hidden_feature_probs[hidden_id];
}
// Store the dot_product result
movie_rating_probs[movie_rating_id] = dot_prod;

// Re-use this thread on another data point:
movie_rating_id += blockDim.x * gridDim.x;
}
}