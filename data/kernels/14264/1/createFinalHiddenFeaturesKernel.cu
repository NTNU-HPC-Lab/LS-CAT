#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createFinalHiddenFeaturesKernel(const float *weights, const float *movie_rating_probs, float* final_hidden_feature_probs, int num_movies, int num_hidden_features) {

// weights[NUM_MOVIES][5][NUM_FEATURES]
// movie_rating_probs[NUM_MOVIES][5]
// final_hidden_feature_probs[NUM_FEATURES]
unsigned int hidden_id = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int movie_id = 0;
unsigned int rating = 0;
float dot_prod; // Temporary, local dot product variable
while (hidden_id < num_hidden_features) {
dot_prod = 0.00; // Initialize the dot product to 0

for (movie_id = 0; movie_id < num_movies; movie_id++) {
for (rating = 0; rating < 5; rating++) {
// Indexing: weights[movie_id][rating][feature_id]
// movie_id - [1, 17771]
// rating - [0, 4]
// hidden_id - [0, 99]
// Do the dot product
dot_prod += weights[movie_id*5*num_hidden_features
+ rating*num_hidden_features
+ hidden_id]
* final_hidden_feature_probs[hidden_id];
}
}
// Store the dot_product result
final_hidden_feature_probs[hidden_id] = dot_prod;

// Re-use this thread on another data point:
hidden_id += blockDim.x * gridDim.x;
}
}