#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecMultiplyReverse(int *A, int *B, int *C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i%2 == 0)
{
C[i] = A[i] + B[i];
}
else if(i%2 != 0)
{
C[i] = A[i] - B[i];
}
}