#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverseArray(int *A, int *B) {
int threadID = threadIdx.x;
int start = (threadID * ArraySize) / 256;
int end = ( ( (threadID + 1 ) * ArraySize) / 256) - 1;
while(end > 0)
{
B[end] = A[start];
end--;
start++;
}
}