#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(int *A, int *B, int *C) {
int i = blockIdx.x*blockDim.x+threadIdx.x;
C[i] = A[i];
}