#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
printf("Block ID X : %d and  Block ID Y: %d\n", blockIdx.x,blockIdx.y);
float Pvalue = 0;
if(row < Width || col < Width){
for(int k=0;k<ncols;k++){
float Melement = Md[row*ncols+k];
float Nelement = Nd[k*ncols+col];
Pvalue += Melement * Nelement;
}
}

Pd[row*ncols+col] = Pvalue;
}