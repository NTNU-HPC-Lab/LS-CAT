#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscale(float4* imagem, int width, int height)
{
const int i = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;

if(i < width * height)
{
float v = 0.3 * imagem[i].x + 0.6 * imagem[i].y + 0.1 * imagem[i].z;
imagem[i] = make_float4(v, v, v, 0);
}
}