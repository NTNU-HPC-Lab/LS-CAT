#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float *A, float *B, float *C) {
int i;

i = blockIdx.x*blockDim.x + threadIdx.x;
C[i] = A[i] + B[i];

}