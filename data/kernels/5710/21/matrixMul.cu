#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(float* A, float* B, float* C, int width)
{
__shared__ float As[TILE_WIDTH] [TILE_WIDTH];
__shared__ float Bs[TILE_WIDTH] [TILE_WIDTH];
int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
float c_val = 0.0f;for(int i = 0; i < width/TILE_WIDTH; i++)
{
As[threadIdx.y][threadIdx.x] = A[row * width + (i * TILE_WIDTH + threadIdx.x)];
Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * width + col ];
__syncthreads();
for(int k = 0; k < TILE_WIDTH; k++)
c_val += As[threadIdx.y][k] * Bs[k][threadIdx.x];__syncthreads();
}
C[row * width + col] = c_val;
}