#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(int *A, int *B, int *C) {
int i = threadIdx.x;
C[i] = A[i] + B[i];
}