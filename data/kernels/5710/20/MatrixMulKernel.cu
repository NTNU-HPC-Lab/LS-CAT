#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {

int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

// Pvalue is used to store the element of the output matrix
// that is computed by the thread

float Pvalue = 0;
for (int k=0; k < ncols; ++k) {
float Melement = Md[row*ncols+k];
float Nelement = Nd[k*ncols+col];
Pvalue += Melement * Nelement;
}

Pd[row*ncols+col] = Pvalue;
}