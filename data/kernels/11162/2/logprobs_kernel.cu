#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void logprobs_kernel(double * corect_logprobs, double * probs, int* y, int size)
{
int i = blockIdx.x;
corect_logprobs[i] = -log(probs[i*size + y[i]]);
}