#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void dhidden_cal_kernel(double * a1,double * dhidden,int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
if (a1[i*size + j] <= 0)
{
dhidden[i*size + j] = 0;
}
}