#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void grads_w1_kernel(double * grads_W1,double * W1,double reg, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
grads_W1[i*size + j] += W1[i*size + j] * reg;
}