#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void x_batch_kernel(double* X_batch, double * X, int * sample_indices, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
X_batch[i*size + j] = X[sample_indices[i] * size + j];
}