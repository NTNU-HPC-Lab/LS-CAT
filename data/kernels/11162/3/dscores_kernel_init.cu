#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void dscores_kernel_init(int * y, double * dscores, int size)
{
int i = blockIdx.x;
dscores[i*size + y[i]] -= 1;
}