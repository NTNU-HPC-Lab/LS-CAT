#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void bfs(const edge *edges, int *vertices, int current_depth) {
int a = blockDim.x * blockIdx.x + threadIdx.x;
int vfirst = edges[a].first;
int dfirst = vertices[vfirst];
int vsecond = edges[a].second;
int dsecond = vertices[vsecond];
if ((dfirst == current_depth) && (dsecond == -1)) {
vertices[vsecond] = dfirst + 1;
}
if ((dfirst == -1) && (dsecond == current_depth)) {
vertices[vfirst] = dsecond + 1;
}
}