#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void dscore_cal_kernel(double * dscores, int num_inputs, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
dscores[i*size + j] /= num_inputs;
}