#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void w1_kernel(double * grads_W1, double * W1, double learning_rate, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
W1[i*size + j] += (-learning_rate * grads_W1[i*size + j]);
}