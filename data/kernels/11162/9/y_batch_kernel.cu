#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void y_batch_kernel(double* y_batch, double * y, int * sample_indices, int size)
{
int i = blockIdx.x;
y_batch[i] = y[sample_indices[i]];
}