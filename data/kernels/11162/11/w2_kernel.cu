#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void w2_kernel(double * grads_W2, double * W2, double learning_rate, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
W2[i*size + j] += (-learning_rate * grads_W2[i*size + j]);
}