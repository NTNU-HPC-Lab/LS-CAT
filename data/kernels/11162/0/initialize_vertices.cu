#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void initialize_vertices(int *vertices, int starting_vertex) {
int v = blockDim.x * blockIdx.x + threadIdx.x;
if (v == starting_vertex) vertices[v] = 0; else vertices[v] = -1;
}