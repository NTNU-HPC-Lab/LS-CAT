#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;



typedef struct edge {
int first, second;
} edges;





__global__ void grads_w2_kernel(double * grads_W2,double * W2,double reg, int size)
{
int i = blockIdx.x;
int j = threadIdx.x;
grads_W2[i*size + j] += W2[i*size + j] * reg;
}