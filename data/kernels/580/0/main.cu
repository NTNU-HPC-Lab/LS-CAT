#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "network_corr.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *templates = NULL;
hipMalloc(&templates, XSIZE*YSIZE);
float *sum_square_template = NULL;
hipMalloc(&sum_square_template, XSIZE*YSIZE);
int *moveout = NULL;
hipMalloc(&moveout, XSIZE*YSIZE);
float *data = NULL;
hipMalloc(&data, XSIZE*YSIZE);
float *weights = NULL;
hipMalloc(&weights, XSIZE*YSIZE);
size_t step = 1;
size_t n_samples_template = 1;
size_t n_samples_data = 1;
size_t n_stations = 1;
size_t n_components = 1;
int chunk_offset = 1;
int chunk_size = XSIZE*YSIZE;
float *cc_mat = NULL;
hipMalloc(&cc_mat, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
network_corr<<<gridBlock,threadBlock>>>(templates,sum_square_template,moveout,data,weights,step,n_samples_template,n_samples_data,n_stations,n_components,chunk_offset,chunk_size,cc_mat);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
network_corr<<<gridBlock,threadBlock>>>(templates,sum_square_template,moveout,data,weights,step,n_samples_template,n_samples_data,n_stations,n_components,chunk_offset,chunk_size,cc_mat);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
network_corr<<<gridBlock,threadBlock>>>(templates,sum_square_template,moveout,data,weights,step,n_samples_template,n_samples_data,n_stations,n_components,chunk_offset,chunk_size,cc_mat);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}