#include "hip/hip_runtime.h"
#include "includes.h"
/*
:copyright:
William B. Frank and Eric Beauce
:license:
GNU General Public License, Version 3
(https://www.gnu.org/licenses/gpl-3.0.en.html)
*/

__global__ void network_corr(float *templates, float *sum_square_template, int *moveout, float *data, float *weights, size_t step, size_t n_samples_template, size_t n_samples_data, size_t n_stations, size_t n_components, int chunk_offset, int chunk_size, float *cc_mat) {

// each thread matches the template to one time in the data
int idx, first_sample_block, first_sample_trace, last_sample_trace; // sample's index
int i, s, c; // counters
int data_offset, templates_offset, sum_square_template_offset, cc_mat_offset;
float numerator, denominator, sum_square_data;
float data_sample;
int t_idx;

//------------------------------------------------
int count_template = (n_samples_template / WARPSIZE + 1) * WARPSIZE;
extern __shared__ float shared[];
float *ss_template = &shared[0];
float *templates_s = &shared[sizeof(float)];
float *data_s = &shared[count_template+sizeof(float)];

// 1 block processes one channel to blockDim.x / step different positions in time
idx = blockIdx.x/n_stations * blockDim.x + chunk_offset;
first_sample_block = idx * step;
s = blockIdx.x % n_stations;

for (c = 0; c < n_components; c++){
if (weights[s * n_components + c] != 0.){
// compute offsets for input variables
cc_mat_offset = (first_sample_block / step + threadIdx.x - chunk_offset) * n_stations * n_components + s * n_components + c;
templates_offset = s * n_samples_template * n_components + c * n_samples_template;
sum_square_template_offset = s * n_components + c;
first_sample_trace = first_sample_block + moveout[s * n_components + c];
last_sample_trace = first_sample_trace + n_samples_template + threadIdx.x * step;
data_offset = s * n_samples_data * n_components + c * n_samples_data + first_sample_trace;

// initialize sums
sum_square_data = 0.0f;
numerator = 0.0f;

// load template and data into shared memory
t_idx = threadIdx.x;
if (t_idx == 0){
ss_template[0] = sum_square_template[sum_square_template_offset];
}
while(t_idx < n_samples_template) {
templates_s[t_idx] = templates[templates_offset + t_idx];
if ((first_sample_trace + t_idx) < n_samples_data) data_s[t_idx] = data[data_offset + t_idx];
t_idx += blockDim.x;
}
while(t_idx < (blockDim.x * step + n_samples_template)){
if ((first_sample_trace + t_idx) < n_samples_data) data_s[t_idx] = data[data_offset + t_idx];
t_idx += blockDim.x;
}

__syncthreads(); // make sure the waveforms are read before keep going

// calculate correlation coefficient
if (last_sample_trace < n_samples_data){
// if not, corresponds to an ill-defined CC with some samples out of the bounds
for(i = 0; i < n_samples_template; i++) {
data_sample = data_s[i + threadIdx.x * step];
numerator += data_sample * templates_s[i];
sum_square_data += data_sample * data_sample;
}
//denominator = sum_square_data * sum_square_template[sum_square_template_offset];
denominator = sum_square_data * ss_template[0];
if (cc_mat_offset < (chunk_size * n_stations * n_components)){
// check that this thread is not ouf of the chunk's bounds
if (denominator > STABILITY_THRESHOLD) cc_mat[cc_mat_offset] = numerator * rsqrtf(denominator);
}
}
}
__syncthreads(); // wait for every thread to finish before leaving the kernel
}
}