#include "hip/hip_runtime.h"
#include "includes.h"
/*
:copyright:
William B. Frank and Eric Beauce
:license:
GNU General Public License, Version 3
(https://www.gnu.org/licenses/gpl-3.0.en.html)
*/

__global__ void sum_cc(float *cc_mat, float *cc_sum, float *weights, int n_stations, int n_components, int n_corr, int chunk_offset, int chunk_size) {

int i, ch;

i = blockIdx.x * blockDim.x + threadIdx.x;
if ( ((i + chunk_offset) < n_corr) & (i < chunk_size) ){
// first condition: check if we are not outside cc_sum's length
// second condition: check if we are not outside the chunk's size
float *cc_mat_offset;

cc_mat_offset = cc_mat + i * n_stations * n_components;
for (ch = 0; ch < (n_stations * n_components); ch++) cc_sum[i] += cc_mat_offset[ch] * weights[ch];
}
}