#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel01 (int N, int M, int P, float *A, float *B, float *C) {
__shared__ float sA[SIZE][SIZE];
__shared__ float sB[SIZE][SIZE];

int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by * SIZE + ty;
int col = bx * SIZE + tx;

float tmp = 0.0;
for (int m=0; m< (P/SIZE); m++) {
sA[ty][tx] = A[row*P + m*SIZE + tx];
sB[ty][tx] = B[col + (m*SIZE + ty)*M];
__syncthreads();
for (int k=0; k<SIZE; k++)
tmp += sA[ty][k] * sB[k][tx];
__syncthreads();
}
C[row*M+col] = tmp;
}