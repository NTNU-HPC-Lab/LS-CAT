#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel11(int N, int M, int P, float *A, float *B, float *C) {

__shared__ float sA[SIZE][SIZE];
__shared__ float sB[SIZE][SIZE];

int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by * SIZE + ty;
int col = bx * SIZE + tx;
int m, k, iter;



float tmp = 0.0;
iter = P%SIZE;
if (iter == 0) {
for (m=0; m < P; m=m+SIZE) {
sA[ty][tx] = A[row*P + m + tx];
sB[ty][tx] = B[col + (m + ty)*M];
__syncthreads();
for (k=0; k<SIZE; k++)
tmp += sA[ty][k] * sB[k][tx];
__syncthreads();
}
}
else {
for (m=0; m < P-iter; m=m+SIZE) {
sA[ty][tx] = A[row*P + m + tx];
sB[ty][tx] = B[col + (m + ty)*M];
__syncthreads();
for (k=0; k<SIZE; k++)
tmp += sA[ty][k] * sB[k][tx];
__syncthreads();
}

if (col < P && row < N) sA[ty][tx] = A[row*P + m + tx];   else sA[ty][tx] = 0.0;
if (row < P && col < M) sB[ty][tx] = B[col + (m + ty)*M]; else sB[ty][tx] = 0.0;
__syncthreads();
for (k=0; k<iter; k++)
tmp += sA[ty][k] * sB[k][tx];
}
if ((row < N) && (col < M)) C[row*M+col] = tmp;

}