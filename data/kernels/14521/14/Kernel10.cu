#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel10(int N, int M, int P, float *A, float *B, float *C) {

__shared__ float sA[SIZE][SIZE];
__shared__ float sB[SIZE][SIZE];

int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by * SIZE + ty;
int col = bx * SIZE + tx;

float tmp = 0.0;
for (int m=0; m < P; m=m+SIZE) {
sA[ty][tx] = A[row*P + m + tx];
sB[ty][tx] = B[col + (m + ty)*M];
__syncthreads();
for (int k=0; k<SIZE; k++)
tmp += sA[ty][k] * sB[k][tx];
__syncthreads();
}
C[row*M+col] = tmp;
}