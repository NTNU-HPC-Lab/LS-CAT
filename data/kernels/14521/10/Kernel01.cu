#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel01 (int N, int M, int P, float *A, float *B, float *C) {

int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if (row < N && col < M) {
float tmp = 0.0;
for (int k=0; k<P; k++)
tmp += A[row*P+k] * B[k*N+col];
C[row*N+col] = tmp;
}
}