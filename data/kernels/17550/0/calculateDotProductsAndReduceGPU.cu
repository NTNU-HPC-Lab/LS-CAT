#include "hip/hip_runtime.h"
#include "includes.h"
#define _USE_MATH_DEFINES



__global__ void calculateDotProductsAndReduceGPU(int *vec1, int *vec2, int *reduced, int numElements)
{
extern __shared__ int sdata[];

int tid = threadIdx.x;

int i = blockDim.x * blockIdx.x + threadIdx.x;

int numThreads = blockDim.x;


sdata[tid] = 0;
sdata[tid + numThreads] = 0;
sdata[tid + (numThreads * 2)] = 0;



if (i < numElements)
{
//Multiplications for dot products

sdata[tid] = vec1[i] * vec2[i];
sdata[tid + numThreads] = vec1[i] * vec1[i];
sdata[tid + (numThreads * 2)] = vec2[i] * vec2[i];
__syncthreads();

//Reduction with sequential addressing with reversed loop and threadID-based indexing
for (int s = blockDim.x/2; s > 0; s>>=1)
{
if (tid < s)
{
sdata[tid] += sdata[tid + s];
sdata[tid + numThreads] += sdata[tid + numThreads + s];
sdata[tid + (numThreads * 2)] += sdata[(tid + (numThreads * 2)) + s];
}
__syncthreads();
}
}

if (tid == 0)
{
atomicAdd(&reduced[0], sdata[tid]);
atomicAdd(&reduced[1], sdata[tid + numThreads]);
atomicAdd(&reduced[2], sdata[tid + (numThreads * 2)]);
}
}