#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectify(unsigned char * original_img, unsigned char* new_img, unsigned int num_thread, unsigned int size) {
for (int i = threadIdx.x; i < size; i = i + num_thread) {
if (original_img[i] < 127)
new_img[i] = 127;
else
new_img[i] = original_img[i];
}
}