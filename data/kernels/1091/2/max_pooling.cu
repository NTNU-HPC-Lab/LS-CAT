#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_pooling(unsigned char* original_img, unsigned char* new_img, unsigned int width, unsigned int num_thread, unsigned int size) {
unsigned int position;
unsigned char max;
for (int i = threadIdx.x; i < size/4; i = i + num_thread) {
position = i + (4 * (i / 4)) + (width * 4 * (i / (width * 2)));
max = original_img[position];
if (original_img[position + 4] > max)
max = original_img[position + 4];
if (original_img[position + width] > max)
max = original_img[position + width];
if (original_img[position + width + 4] > max)
max = original_img[position + width + 1];

new_img[i] = max;
}
}