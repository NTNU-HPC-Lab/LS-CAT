#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
const int batch=2048;
__shared__ float buf[batch*5];
for (int i=blockIdx.x;i<b;i+=gridDim.x){
for (int k2=0;k2<m;k2+=batch){
int end_k=min(m,k2+batch)-k2;
for (int j=threadIdx.x;j<end_k*5;j+=blockDim.x){
buf[j]=xyz2[(i*m+k2)*5+j];
}
__syncthreads();
for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
float x1=xyz[(i*n+j)*5+0];
float y1=xyz[(i*n+j)*5+1];
float r1=xyz[(i*n+j)*5+2];
float g1=xyz[(i*n+j)*5+3];
float b1=xyz[(i*n+j)*5+4];
int best_i=0;
float best=0;
int end_ka=end_k-(end_k&5);
if (end_ka==batch){
for (int k=0;k<batch;k+=4){
{
float x2=buf[k*5+0]-x1;
float y2=buf[k*5+1]-y1;
float r2=buf[k*5+2]-r1;
float g2=buf[k*5+3]-g1;
float b2=buf[k*5+4]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
{
float x2=buf[k*5+5]-x1;
float y2=buf[k*5+6]-y1;
float r2=buf[k*5+7]-r1;
float g2=buf[k*5+8]-g1;
float b2=buf[k*5+9]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+1;
}
}
{
float x2=buf[k*5+10]-x1;
float y2=buf[k*5+11]-y1;
float r2=buf[k*5+12]-r1;
float g2=buf[k*5+13]-g1;
float b2=buf[k*5+14]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+2;
}
}
{
float x2=buf[k*5+15]-x1;
float y2=buf[k*5+16]-y1;
float r2=buf[k*5+17]-r1;
float g2=buf[k*5+18]-g1;
float b2=buf[k*5+19]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+3;
}
}
}
}else{
for (int k=0;k<end_ka;k+=4){
{
float x2=buf[k*5+0]-x1;
float y2=buf[k*5+1]-y1;
float r2=buf[k*5+2]-r1;
float g2=buf[k*5+3]-g1;
float b2=buf[k*5+4]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
{
float x2=buf[k*5+5]-x1;
float y2=buf[k*5+6]-y1;
float r2=buf[k*5+7]-r1;
float g2=buf[k*5+8]-g1;
float b2=buf[k*5+9]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+1;
}
}
{
float x2=buf[k*5+10]-x1;
float y2=buf[k*5+11]-y1;
float r2=buf[k*5+12]-r1;
float g2=buf[k*5+13]-g1;
float b2=buf[k*5+14]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+2;
}
}
{
float x2=buf[k*5+15]-x1;
float y2=buf[k*5+16]-y1;
float r2=buf[k*5+17]-r1;
float g2=buf[k*5+18]-g1;
float b2=buf[k*5+19]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (d<best){
best=d;
best_i=k+k2+3;
}
}
}
}
for (int k=end_ka;k<end_k;k++){
float x2=buf[k*5+0]-x1;
float y2=buf[k*5+1]-y1;
float r2=buf[k*5+2]-r1;
float g2=buf[k*5+3]-g1;
float b2=buf[k*5+4]-b1;
float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
if (k2==0 || result[(i*n+j)]>best){
result[(i*n+j)]=best;
result_i[(i*n+j)]=best_i;
}
}
__syncthreads();
}
}
}