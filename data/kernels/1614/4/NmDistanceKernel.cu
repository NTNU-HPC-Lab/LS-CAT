#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
const int batch=512;
__shared__ float buf[batch*2];
for (int i=blockIdx.x;i<b;i+=gridDim.x){
for (int k2=0;k2<m;k2+=batch){
int end_k=min(m,k2+batch)-k2;
for (int j=threadIdx.x;j<end_k*2;j+=blockDim.x){
buf[j]=xyz2[(i*m+k2)*2+j];
}
__syncthreads();
for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
float x1=xyz[(i*n+j)*2+0];
float y1=xyz[(i*n+j)*2+1];
int best_i=0;
float best=0;
int end_ka=end_k-(end_k&2);
if (end_ka==batch){
for (int k=0;k<batch;k+=4){
{
float x2=buf[k*2+0]-x1;
float y2=buf[k*2+1]-y1;
float d=x2*x2+y2*y2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
{
float x2=buf[k*2+2]-x1;
float y2=buf[k*2+3]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+1;
}
}
{
float x2=buf[k*2+4]-x1;
float y2=buf[k*2+5]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+2;
}
}
{
float x2=buf[k*2+6]-x1;
float y2=buf[k*2+7]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+3;
}
}
}
}else{
for (int k=0;k<end_ka;k+=4){
{
float x2=buf[k*2+0]-x1;
float y2=buf[k*2+1]-y1;
float d=x2*x2+y2*y2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
{
float x2=buf[k*2+2]-x1;
float y2=buf[k*2+3]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+1;
}
}
{
float x2=buf[k*2+4]-x1;
float y2=buf[k*2+5]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+2;
}
}
{
float x2=buf[k*2+6]-x1;
float y2=buf[k*2+7]-y1;
float d=x2*x2+y2*y2;
if (d<best){
best=d;
best_i=k+k2+3;
}
}
}
}
for (int k=end_ka;k<end_k;k++){
float x2=buf[k*2+0]-x1;
float y2=buf[k*2+1]-y1;
float d=x2*x2+y2*y2;
if (k==0 || d<best){
best=d;
best_i=k+k2;
}
}
if (k2==0 || result[(i*n+j)]>best){
result[(i*n+j)]=best;
result_i[(i*n+j)]=best_i;
}
}
__syncthreads();
}
}
}