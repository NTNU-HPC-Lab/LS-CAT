#include "hip/hip_runtime.h"
#include "includes.h"






__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
for (int i=blockIdx.x;i<b;i+=gridDim.x){
for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
float x1=xyz1[(i*n+j)*3+0];
float y1=xyz1[(i*n+j)*3+1];
float z1=xyz1[(i*n+j)*3+2];
int j2=idx1[i*n+j];
float x2=xyz2[(i*m+j2)*3+0];
float y2=xyz2[(i*m+j2)*3+1];
float z2=xyz2[(i*m+j2)*3+2];
float g=grad_dist1[i*n+j]*2;
atomicAdd(&(grad_xyz1[(i*n+j)*3+0]),g*(x1-x2));
atomicAdd(&(grad_xyz1[(i*n+j)*3+1]),g*(y1-y2));
atomicAdd(&(grad_xyz1[(i*n+j)*3+2]),g*(z1-z2));
atomicAdd(&(grad_xyz2[(i*m+j2)*3+0]),-(g*(x1-x2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*3+1]),-(g*(y1-y2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*3+2]),-(g*(z1-z2)));
}
}
}