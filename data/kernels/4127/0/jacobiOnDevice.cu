#include "hip/hip_runtime.h"
#include "includes.h"

static char* program_name;

// Usage
__global__ void jacobiOnDevice(float* x_next, float* A, float* x_now, float* b, int Ni, int Nj)
{
float sigma = 0.0;
int idx = threadIdx.x;
for (int j=0; j<Nj; j++)
{
if (idx != j)
sigma += A[idx*Nj + j] * x_now[j];
}
x_next[idx] = (b[idx] - sigma) / A[idx*Nj + idx];
}