#include "hip/hip_runtime.h"
#include "includes.h"

static char* program_name;

// Usage
__global__ void jacobiOptimizedOnDevice(float* x_next, float* A, float* x_now, float* b, int Ni, int Nj)
{
// Optimization step 1: tiling
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < Ni)
{
float sigma = 0.0;

// Optimization step 2: store index in register
// Multiplication is not executed in every iteration.
int idx_Ai = idx*Nj;

// Tried to use prefetching, but then the result is terribly wrong and I don't know why..
/*
float curr_A = A[idx_Ai];
float nxt_A;
//printf("idx=%d\n",idx);
for (int j=0; j<Nj-1; j++)
{
if (idx != j)
nxt_A = A[idx_Ai + j + 1];
sigma += curr_A * x_now[j];
//sigma += A[idx_Ai + j] * x_now[j];
curr_A = nxt_A;
//printf("curr_A=%f\n",curr_A);
}
if (idx != Nj-1)
sigma += nxt_A * x_now[Nj-1];
x_next[idx] = (b[idx] - sigma) / A[idx_Ai + idx];
*/

for (int j=0; j<Nj; j++)
if (idx != j)
sigma += A[idx_Ai + j] * x_now[j];

// Tried to use loop-ennrolling, but also here this gives a wrong result..
/*
for (int j=0; j<Nj/4; j+=4)
{
if (idx != j)
{
sigma += A[idx_Ai + j] * x_now[j];
}
if (idx != j+1)
{
sigma += A[idx_Ai + j+1] * x_now[j+1];
}
if (idx != j+2)
{
sigma += A[idx_Ai + j+2] * x_now[j+2];
}
if (idx != j+3)
{
sigma += A[idx_Ai + j+3] * x_now[j+3];
}
}*/

x_next[idx] = (b[idx] - sigma) / A[idx_Ai + idx];
}
}