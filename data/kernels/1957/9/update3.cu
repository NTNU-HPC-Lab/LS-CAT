#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
*rho_out = 1.0f / *yDotS;

if (*yDotY > 1e-5)
*H0_out = *yDotS / *yDotY;
}