#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convert_float2bgr(float* annd, unsigned char* bgr, int w, int h, float minval, float maxval)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x < w && y < h)
{
int id = y * w + x;
int err = max(min((annd[id] - minval) / (maxval - minval), 1.f), 0.f) * 255.f;

bgr[id] = err;
}
}