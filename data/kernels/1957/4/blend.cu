#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blend(float *cmap, float* oldd, float* newd, float weight,int * params)
{
int ax = blockIdx.x*blockDim.x + threadIdx.x;
int ay = blockIdx.y*blockDim.y + threadIdx.y;

int ch = params[0];
int ah = params[1];
int aw = params[2];

int slice_a = ah * aw;
int pitch_a = aw;

float thre = 0.05;

if (ax < aw&& ay < ah)
{
float fa = cmap[ay*pitch_a + ax];
if (fa < thre)
fa = 0.0f;
else fa = weight;
for (int i = 0; i < ch; i++)
{

newd[i*slice_a + ay*pitch_a + ax] = oldd[i*slice_a + ay*pitch_a + ax]* fa + newd[i*slice_a + ay*pitch_a + ax] * (1.0-fa);
}
}
}