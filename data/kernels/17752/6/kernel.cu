#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
float diff, squareSum;
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < n) {
squareSum = 0;
for (int i = 0; i < k; i++) {
diff = sub_searchPoints[i] - referencePoints[k * tid + i];
squareSum += (diff * diff);
}
dist[tid] = squareSum;
}
}