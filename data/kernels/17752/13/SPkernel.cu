#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SPkernel(int k, int m, int n, float* searchPoints, float* referencePoints, int* indices)
{
int minIndex;
float minSquareSum, diff, squareSum;
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < m) {
minSquareSum = -1;
// Iterate over all reference points
for (int nInd = 0; nInd < n; nInd++) {
squareSum = 0;
for (int kInd = 0; kInd < k; kInd++) {
diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
squareSum += (diff * diff);
}
if (minSquareSum < 0 || squareSum < minSquareSum) {
minSquareSum = squareSum;
minIndex = nInd;
}
}
indices[tid] = minIndex;
}
}