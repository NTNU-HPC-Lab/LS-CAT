#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel4(int k, int n, int bias, float* searchPoints, float* referencePoints, float* dist)
{
float diff, squareSum;
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < n) {
squareSum = 0;
for (int i = 0; i < k; i++) {
diff = searchPoints[k * bias + i] - referencePoints[k * tid + i];
squareSum += (diff * diff);
}
dist[bias * n + tid] = squareSum;
}
}