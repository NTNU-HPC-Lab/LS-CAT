#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 1e-4

__global__ void reconstruction_best_kernel( float *input, float *filtered_affine_model, float *filtered_best_output, int h, int w )
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
if (id < size) {
double out1 =
input[id + 2*size] * filtered_affine_model[id*12 + 0] + // A[0][0] +
input[id + size]   * filtered_affine_model[id*12 + 1] + // A[0][1] +
input[id]          * filtered_affine_model[id*12 + 2] + // A[0][2] +
filtered_affine_model[id*12 + 3]; //A[0][3];
double out2 =
input[id + 2*size] * filtered_affine_model[id*12 + 4] + //A[1][0] +
input[id + size]   * filtered_affine_model[id*12 + 5] + //A[1][1] +
input[id]          * filtered_affine_model[id*12 + 6] + //A[1][2] +
filtered_affine_model[id*12 + 7]; //A[1][3];
double out3 =
input[id + 2*size] * filtered_affine_model[id*12 + 8] + //A[2][0] +
input[id + size]   * filtered_affine_model[id*12 + 9] + //A[2][1] +
input[id]          * filtered_affine_model[id*12 + 10] + //A[2][2] +
filtered_affine_model[id*12 + 11]; // A[2][3];

filtered_best_output[id] = out1;
filtered_best_output[id + size] = out2;
filtered_best_output[id + 2*size] = out3;
}
return ;
}