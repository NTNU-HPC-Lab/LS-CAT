#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void squaresSum(int *data, int *sum, clock_t *time)
{
int sum_t = 0;
clock_t start = clock();
for (int i = 0; i < DATA_SIZE; ++i) {
sum_t += data[i] * data[i];
}
*sum = sum_t;
*time = clock() - start;
}