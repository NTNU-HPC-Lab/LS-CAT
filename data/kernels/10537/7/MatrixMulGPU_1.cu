#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulGPU_1(float *c, const float *a, const float *b, unsigned int WA, unsigned int WB) {
float sum = 0;
//找出该线程所在的行和列
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

//线程Thread(row, col)负责计算C(row, col)
for (int i = 0; i < WB; ++i) {
sum += a[row * WA + i] * b[i * WB + col];
}

c[row * WB + col] = sum;
}