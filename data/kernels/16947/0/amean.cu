#include "hip/hip_runtime.h"
#include "includes.h"
#define N 1024*4
// Device Kernel

//host Function
__global__ void amean(float *A, float *S)
{
//holds intermediates in shared memory reduction
__shared__ int sdata[N];

int tid=threadIdx.x;
int i = blockIdx.x * blockDim.x + threadIdx.x;
sdata[tid]=A[i];
__syncthreads();

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

if(tid==0)
S[blockIdx.x]=sdata[0];

}