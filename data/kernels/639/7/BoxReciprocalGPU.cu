#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BoxReciprocalGPU(double *gpu_prefact, double *gpu_sumRnew, double *gpu_sumInew, double *gpu_energyRecip, int imageSize)
{
int threadID = blockIdx.x * blockDim.x + threadIdx.x;
if(threadID >= imageSize)
return;

gpu_energyRecip[threadID] = ((gpu_sumRnew[threadID] * gpu_sumRnew[threadID] +
gpu_sumInew[threadID] * gpu_sumInew[threadID]) *
gpu_prefact[threadID]);
}