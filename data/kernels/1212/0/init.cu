#include "hip/hip_runtime.h"
#include "includes.h"


// CUDA kernel to add elements of two arrays
__global__
__global__ void init(int n, float *x, float *y) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride) {
x[i] = 1.0f;
y[i] = 2.0f;
}
}