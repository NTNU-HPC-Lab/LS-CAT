#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void predicate_kernel(float *d_predicates, float *d_input, int length)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;

if (idx >= length) return;

d_predicates[idx] = d_input[idx] > FLT_ZERO;
}