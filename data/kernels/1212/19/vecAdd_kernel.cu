#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = 0; i < 500; i++)
c[idx] = a[idx] + b[idx];
}