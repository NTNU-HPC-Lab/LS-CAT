#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

extern __shared__ float s_data[];

// cumulates input with grid-stride loop and save to share memory
float input[NUM_LOAD] = {0.f};
for (int i = idx_x; i < size; i += blockDim.x * gridDim.x * NUM_LOAD)
{
for (int step = 0; step < NUM_LOAD; step++)
input[step] += (i + step * blockDim.x * gridDim.x < size) ? g_in[i + step * blockDim.x * gridDim.x] : 0.f;
}
for (int i = 1; i < NUM_LOAD; i++)
input[0] += input[i];
s_data[threadIdx.x] = input[0];

__syncthreads();

// do reduction
for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (threadIdx.x < stride)
s_data[threadIdx.x] += s_data[threadIdx.x + stride];

__syncthreads();
}

if (threadIdx.x == 0) {
g_out[blockIdx.x] = s_data[0];
}
}