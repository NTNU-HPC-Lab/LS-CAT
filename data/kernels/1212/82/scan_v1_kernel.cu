#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_v1_kernel(float *d_output, float *d_input, int length)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;

float element = 0.f;
for (int offset = 0; offset < length; offset++) {
if (idx - offset >= 0)
element += d_input[idx - offset];
}
d_output[idx] = element;
}