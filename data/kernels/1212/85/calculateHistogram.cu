#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateHistogram(unsigned int *imageHistogram, unsigned int width, unsigned int height, hipTextureObject_t texObj)
{
const unsigned int tidX = blockIdx.x*blockDim.x + threadIdx.x;
const unsigned int tidY = blockIdx.y*blockDim.y + threadIdx.y;

const unsigned int localId = threadIdx.y*blockDim.x+threadIdx.x;
const unsigned int histStartIndex = (blockIdx.y*gridDim.x+blockIdx.x) * 256;

__shared__ unsigned int histo_private[256];

if(localId <256)
histo_private[localId] = 0;
__syncthreads();

// Step 4: Read the texture memory from your texture reference in CUDA Kernel
unsigned char imageData =  tex2D<unsigned char>(texObj,(float)(tidX),(float)(tidY));
atomicAdd(&(histo_private[imageData]), 1);

__syncthreads();

if(localId < 256)
imageHistogram[histStartIndex+localId] = histo_private[localId];

}