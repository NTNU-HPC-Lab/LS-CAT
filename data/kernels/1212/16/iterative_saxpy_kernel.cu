#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iterative_saxpy_kernel(float *y, const float* x, const float alpha, const float beta, int n_loop)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = 0; i < n_loop; i++)
y[idx] = alpha * x[idx] + beta;
}