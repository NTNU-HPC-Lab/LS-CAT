#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fmaf_kernel(float *d_x, float *d_y, float *d_z, int size)
{
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;

for (int i = idx_x; i < size; i += stride) {
d_z[i] = fmaf(d_x[i], d_y[i], 0.f);
}
}