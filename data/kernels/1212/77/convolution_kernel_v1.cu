#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_kernel_v1(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filter_size)
{
int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

float result = 0.f;
for (int filter_row = -filter_size / 2; filter_row <= filter_size / 2; ++filter_row)
{
for (int filter_col = -filter_size / 2; filter_col <= filter_size / 2; ++filter_col)
{
// Find the global position to apply the given filter
int image_row = idx_y + filter_row;
int image_col = idx_x + filter_col;

float image_value = (image_row >= 0 && image_row < num_row && image_col >= 0 && image_col < num_col) ?
d_input[image_row * num_col + image_col] : 0.f;
float filter_value = d_filter[(filter_row + filter_size / 2) * filter_size + filter_col + filter_size / 2];

result += image_value * filter_value;
}
}

d_output[idx_y * num_col + idx_x] = result;
}