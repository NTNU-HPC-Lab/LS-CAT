#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

extern __shared__ float s_data[];

// cumulates input with grid-stride loop and save to share memory
float input = 0.f;
for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
input += g_in[i];
s_data[threadIdx.x] = input;

__syncthreads();

// do reduction
for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (threadIdx.x < stride)
s_data[threadIdx.x] += s_data[threadIdx.x + stride];

__syncthreads();
}

if (threadIdx.x == 0) {
g_out[blockIdx.x] = s_data[0];
}
}