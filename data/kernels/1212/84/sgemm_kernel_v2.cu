#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel_v2(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
int bid_x = blockIdx.x * blockDim.x;
int bid_y = blockIdx.y * blockDim.y;
int tid_x = threadIdx.x;
int tid_y = threadIdx.y;

float element_c = 0.f;
__shared__ float s_tile_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float s_tile_B[BLOCK_DIM][BLOCK_DIM];

// forward tile with tile size in matrix A
for (int k = 0; k < K; k += BLOCK_DIM)
{
s_tile_A[tid_y][tid_x] = A[ (bid_y + tid_y) * K + tid_x + k ]; // Get sub-matrix from A
s_tile_B[tid_y][tid_x] = B[ (k*BLOCK_DIM + tid_y) * N + bid_x + tid_x ]; // Get sub-matrix from B

__syncthreads();

// compute gemm operation with tiles
for (int e = 0; e < BLOCK_DIM; e++)
element_c += s_tile_A[tid_y][e] * s_tile_B[e][tid_x];

__syncthreads();
}

C[(bid_y + tid_y) * N + (bid_x + tid_x)] = \
alpha * element_c + beta * C[(bid_y + tid_y) * N + (bid_x + tid_x)];
}