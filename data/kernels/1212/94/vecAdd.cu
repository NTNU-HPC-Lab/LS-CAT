#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < len)
out[i] = in1[i] + in2[i];
}