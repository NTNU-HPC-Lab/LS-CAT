#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

extern __shared__ float s_data[];

s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;

__syncthreads();

// do reduction
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
{
// thread synchronous reduction
if ( (idx_x % (stride * 2)) == 0 )
s_data[threadIdx.x] += s_data[threadIdx.x + stride];

__syncthreads();
}

if (threadIdx.x == 0)
d_out[blockIdx.x] = s_data[0];
}