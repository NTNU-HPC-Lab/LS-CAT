#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_updateweights_fl(int N, float *wt, float *x, float *q, float nu){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<N) {
wt[tid]=((nu+1.0f)/(nu+x[tid]*x[tid]));
q[tid]=wt[tid]-logf(wt[tid]); /* so that its +ve */
}
}