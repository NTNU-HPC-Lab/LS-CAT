#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard_sum(int N, double *y, double *x, double *w){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only N threads */
if (tid<N) {
y[tid]+=x[tid]*w[tid];
}
}