#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setweights_fl(int N, float *wt, float alpha){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<N) {
wt[tid]=alpha;
}
}