#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_diagdiv_fl(int M, float eps, float *y, float *x){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<M) {
if (x[tid]>eps) {
y[tid]=y[tid]/x[tid];
} else {
y[tid]=0.0f;
}
}
}