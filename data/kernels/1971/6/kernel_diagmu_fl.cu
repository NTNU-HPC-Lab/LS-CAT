#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_diagmu_fl(int M, float *A,float mu){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<M) {
A[tid*(M+1)]=A[tid*(M+1)]+mu;
}
}