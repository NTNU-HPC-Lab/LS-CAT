#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard_fl(int N, float *wt, float *x){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<N) {
x[tid]*=wt[tid];
}
}