#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setweights(int N, double *wt, double alpha){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only N threads */
if (tid<N) {
wt[tid]=alpha;
}
}