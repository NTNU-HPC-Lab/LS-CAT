#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double digamma(double x) {
double result = 0.0, xx, xx2, xx4;
for ( ; x < 7.0; ++x) { /* reduce x till x<7 */
result -= 1.0/x;
}
x -= 1.0/2.0;
xx = 1.0/x;
xx2 = xx*xx;
xx4 = xx2*xx2;
result += log(x)+(1./24.)*xx2-(7.0/960.0)*xx4+(31.0/8064.0)*xx4*xx2-(127.0/30720.0)*xx4*xx4;
return result;
}
__global__ void kernel_evaluatenu(int Nd, double qsum, double *q, double deltanu,double nulow) {
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid<Nd) {
double thisnu=(nulow+((double)tid)*deltanu);
double dgm=digamma(thisnu*0.5+0.5);
q[tid]=dgm-log((thisnu+1.0)*0.5); /* psi((nu+1)/2)-log((nu+1)/2) */
dgm=digamma(thisnu*0.5);
q[tid]+=-dgm+log((thisnu)*0.5); /* -psi((nu)/2)+log((nu)/2) */
q[tid]+=-qsum+1.0; /* -(-sum(ln(w_i))/N+sum(w_i)/N)+1 */
}
}