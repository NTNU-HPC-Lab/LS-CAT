#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float digamma_fl(float x) {
float result = 0.0f, xx, xx2, xx4;
for ( ; x < 7.0f; ++x) { /* reduce x till x<7 */
result -= 1.0f/x;
}
x -= 1.0f/2.0f;
xx = 1.0f/x;
xx2 = xx*xx;
xx4 = xx2*xx2;
result += logf(x)+(1.0f/24.0f)*xx2-(7.0f/960.0f)*xx4+(31.0f/8064.0f)*xx4*xx2-(127.0f/30720.0f)*xx4*xx4;
return result;
}
__device__ double digamma(double x) {
double result = 0.0, xx, xx2, xx4;
for ( ; x < 7.0; ++x) { /* reduce x till x<7 */
result -= 1.0/x;
}
x -= 1.0/2.0;
xx = 1.0/x;
xx2 = xx*xx;
xx4 = xx2*xx2;
result += log(x)+(1./24.)*xx2-(7.0/960.0)*xx4+(31.0/8064.0)*xx4*xx2-(127.0/30720.0)*xx4*xx4;
return result;
}
__global__ void kernel_evaluatenu_fl(int Nd, float qsum, float *q, float deltanu,float nulow) {
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid<Nd) {
float thisnu=(nulow+((float)tid)*deltanu);
float dgm=digamma_fl(thisnu*0.5f+0.5f);
q[tid]=dgm-logf((thisnu+1.0f)*0.5f); /* psi((nu+1)/2)-log((nu+1)/2) */
dgm=digamma_fl(thisnu*0.5f);
q[tid]+=-dgm+logf((thisnu)*0.5f); /* -psi((nu)/2)+log((nu)/2) */
q[tid]+=-qsum+1.0f; /* -(-sum(ln(w_i))/N+sum(w_i)/N)+1 */
}
}