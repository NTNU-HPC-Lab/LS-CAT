#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_updateweights(int N, double *wt, double *x, double *q, double nu){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only N threads */
if (tid<N) {
wt[tid]=((nu+1.0)/(nu+x[tid]*x[tid]));
q[tid]=wt[tid]-log(wt[tid]); /* so that its +ve */
}
}