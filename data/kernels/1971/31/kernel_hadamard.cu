#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard(int N, double *wt, double *x){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only N threads */
if (tid<N) {
x[tid]*=wt[tid];
}
}