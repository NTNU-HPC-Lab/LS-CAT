#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sqrtweights(int N, double *wt){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only N threads */
if (tid<N) {
wt[tid]=sqrt(wt[tid]);
}
}