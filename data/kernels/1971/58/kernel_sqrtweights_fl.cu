#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sqrtweights_fl(int N, float *wt){
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* make sure to use only M threads */
if (tid<N) {
wt[tid]=sqrtf(wt[tid]);
}
}