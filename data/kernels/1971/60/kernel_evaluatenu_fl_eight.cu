#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float digamma_fl(float x) {
float result = 0.0f, xx, xx2, xx4;
for ( ; x < 7.0f; ++x) { /* reduce x till x<7 */
result -= 1.0f/x;
}
x -= 1.0f/2.0f;
xx = 1.0f/x;
xx2 = xx*xx;
xx4 = xx2*xx2;
result += logf(x)+(1.0f/24.0f)*xx2-(7.0f/960.0f)*xx4+(31.0f/8064.0f)*xx4*xx2-(127.0f/30720.0f)*xx4*xx4;
return result;
}
__device__ double digamma(double x) {
double result = 0.0, xx, xx2, xx4;
for ( ; x < 7.0; ++x) { /* reduce x till x<7 */
result -= 1.0/x;
}
x -= 1.0/2.0;
xx = 1.0/x;
xx2 = xx*xx;
xx4 = xx2*xx2;
result += log(x)+(1./24.)*xx2-(7.0/960.0)*xx4+(31.0/8064.0)*xx4*xx2-(127.0/30720.0)*xx4*xx4;
return result;
}
__global__ void kernel_evaluatenu_fl_eight(int Nd, float qsum, float *q, float deltanu,float nulow, float nu0) {
unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
/* each block calculte  psi((nu+8)/2)-log((nu+8)/2) */
/* actually p=2, so psi((nu+2)/2)-log((nu+2)/2) */
float dgm0;
if (threadIdx.x==0) {
dgm0=digamma_fl(nu0*0.5f+1.0f);
dgm0=dgm0-logf((nu0+2.0f)*0.5f); /* psi((nu0+8)/2)-log((nu0+8)/2) */
}
__syncthreads();
if (tid<Nd) {
float thisnu=(nulow+((float)tid)*deltanu);
q[tid]=dgm0; /* psi((nu0+8)/2)-log((nu0+8)/2) */
float dgm=digamma_fl(thisnu*0.5f);
q[tid]+=-dgm+logf((thisnu)*0.5f); /* -psi((nu)/2)+log((nu)/2) */
q[tid]+=-qsum+1.0f; /* -(-sum(ln(w_i))/N+sum(w_i)/N)+1 */
}
}