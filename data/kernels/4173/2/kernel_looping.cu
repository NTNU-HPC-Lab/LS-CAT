#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_looping(float *point, unsigned int num) {
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

for (int iloop = 0; iloop < NLOOPS; ++iloop) {
for (size_t offset = idx; offset < num; offset += gridDim.x * blockDim.x) {
point[offset] += 1;
}
}
}