#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(const float *input, float *output, int numElements) {
float val = 0.f;
for (int i = 0; i < numElements; ++i) {
val += input[i];
}
*output = val;
}