#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
int myId = threadIdx.x + blockDim.x * blockIdx.x;
int myItem = d_in[myId];
int myBin = myItem % BIN_COUNT;
d_bins[myBin]++;
}