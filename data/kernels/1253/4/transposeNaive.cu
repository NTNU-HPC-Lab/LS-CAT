#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaive(float *odata, float *idata, int width, int height)
{
int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

int index_in  = xIndex + width * yIndex;
int index_out = yIndex + height * xIndex;

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
odata[index_out+i] = idata[index_in+i*width];
}
}