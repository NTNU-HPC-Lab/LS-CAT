#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void displayAttributeValues() {

printf("\nthreadIdx.x : %d,  threadIdx.y : %d,  threadIdx.z : %d,"
"  blockIdx.x : %d,  blockIdx.y : %d,  blockIdx.z : %d,"
"  blockDim.x : %d,  blockDim.y : %d,  blockDim.z : %d,"
"  gridDim.x : %d,  gridDim.y : %d,  gridDim.z : %d\n",
threadIdx.x,threadIdx.y,threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z,
blockDim.x, blockDim.y, blockDim.z,
gridDim.x, gridDim.y, gridDim.z);

}