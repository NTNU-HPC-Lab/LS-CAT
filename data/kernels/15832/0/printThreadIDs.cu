#include "hip/hip_runtime.h"
#include "includes.h"
// ******************************************************************************************************
// PURPOSE		:	Print thread IDs for the 256 threads of a 2D configuration (16 * 16)				*
// LANGUAGE		:		CUDA C / CUDA C++																*
// ASSUMPTIONS	:	2D Configuration 16 threads in each x & y directions with thread block of (8*8)		*
//					threadIdx.z value will be zero since it is 2D configuration							*
// DATE			:	23 March 2020																		*
// AUTHOR		:	Vaibhav BENDRE 																		*
//					vaibhav.bendre7520@gmail.com														*
// ******************************************************************************************************




__global__ void printThreadIDs() {

printf("\n threadIdx.x : %d,   threadIdx.y :  %d ",threadIdx.x,threadIdx.y);

}