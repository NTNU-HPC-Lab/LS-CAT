#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void update_inverse_cuda (float *Ainv, float *u, int N, int rowstride, int k)
{
__shared__ float A_k[NMAX], u_shared[NMAX], Ainv_u[NMAX], Ainv_shared[NMAX];
A_k[threadIdx.x] = Ainv[k*rowstride+threadIdx.x];
u_shared[threadIdx.x] = u[threadIdx.x];

// First, compute k'th element of Ainv_u
Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * A_k[threadIdx.x];
__syncthreads();
for (int n=N>>1; n>0; n = n>>1) {
float a;
if (threadIdx.x < n)
a = Ainv_u[2*threadIdx.x] + Ainv_u[2*threadIdx.x+1];
__syncthreads();
Ainv_u[threadIdx.x] = a;
__syncthreads();
}
float prefact = -1.0f/(1.0f + Ainv_u[0]);

for (int row=0; row<N; row++) {
Ainv_shared[threadIdx.x] = Ainv[row*rowstride+threadIdx.x];
__syncthreads();
Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * Ainv_shared[threadIdx.x];
for (int n=N>>1; n>0; n = n>>1) {
float a;
if (threadIdx.x < n)
a = Ainv_u[2*threadIdx.x] + Ainv_u[2*threadIdx.x+1];
__syncthreads();
Ainv_u[threadIdx.x] = a;
__syncthreads();
}
__syncthreads();
// Now Ainv_u[0] has the row'th element of Ainv_u.
Ainv[row*rowstride + threadIdx.x] =
Ainv_shared[threadIdx.x] + prefact*Ainv_u[0]*A_k[threadIdx.x];
}

}