#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRgb2CIELab(const hipTextureObject_t texFrameBGRA, hipSurfaceObject_t surfFrameLab, int width, int height) {

int px = blockIdx.x*blockDim.x + threadIdx.x;
int py = blockIdx.y*blockDim.y + threadIdx.y;

if (px<width && py<height) {
uchar4 nPixel = tex2D<uchar4>(texFrameBGRA, px, py);//inputImg[offset];

float _b = (float)nPixel.x / 255.0;
float _g = (float)nPixel.y / 255.0;
float _r = (float)nPixel.z / 255.0;

float x = _r * 0.412453 + _g * 0.357580 + _b * 0.180423;
float y = _r * 0.212671 + _g * 0.715160 + _b * 0.072169;
float z = _r * 0.019334 + _g * 0.119193 + _b * 0.950227;

x /= 0.950456;
float y3 = exp(log(y) / 3.0);
z /= 1.088754;

float l, a, b;

x = x > 0.008856 ? exp(log(x) / 3.0) : (7.787 * x + 0.13793);
y = y > 0.008856 ? y3 : 7.787 * y + 0.13793;
z = z > 0.008856 ? z /= exp(log(z) / 3.0) : (7.787 * z + 0.13793);

l = y > 0.008856 ? (116.0 * y3 - 16.0) : 903.3 * y;
a = (x - y) * 500.0;
b = (y - z) * 200.0;

float4 fPixel;
fPixel.x = l;
fPixel.y = a;
fPixel.z = b;
fPixel.w = 0;

fPixel.x = (float)nPixel.x;
fPixel.y = (float)nPixel.y;
fPixel.z = (float)nPixel.z;
fPixel.w = (float)nPixel.w;

surf2Dwrite(fPixel, surfFrameLab, px * 16, py);
}
}