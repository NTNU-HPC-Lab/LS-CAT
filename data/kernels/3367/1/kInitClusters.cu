#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kInitClusters(const hipSurfaceObject_t surfFrameLab, float* clusters, int width, int height, int nSpxPerRow, int nSpxPerCol) {
int centroidIdx = blockIdx.x*blockDim.x + threadIdx.x;
int nSpx = nSpxPerCol*nSpxPerRow;

if (centroidIdx<nSpx){
int wSpx = width / nSpxPerRow;
int hSpx = height / nSpxPerCol;

int i = centroidIdx / nSpxPerRow;
int j = centroidIdx%nSpxPerRow;

int x = j*wSpx + wSpx / 2;
int y = i*hSpx + hSpx / 2;

float4 color;
surf2Dread(&color, surfFrameLab, x * 16, y);
clusters[centroidIdx] = color.x;
clusters[centroidIdx + nSpx] = color.y;
clusters[centroidIdx + 2 * nSpx] = color.z;
clusters[centroidIdx + 3 * nSpx] = x;
clusters[centroidIdx + 4 * nSpx] = y;
}
}