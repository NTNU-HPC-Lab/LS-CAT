#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PositiveDefiniteKernel( char *hessian_pd, float *hessian, int imageW, int imageH, int imageD )
{
const int baseX = blockIdx.x * PD_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * PD_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * PD_BLOCKDIM_Z + threadIdx.z;
const int size = imageW * imageH * imageD;
const int idx = (baseZ * imageH + baseY) * imageW + baseX;

float xx = hessian[idx];
float xy = hessian[idx + size];
float xz = hessian[idx + size*2];
float yy = hessian[idx + size*3];
float yz = hessian[idx + size*4];
float zz = hessian[idx + size*5];

// Sylvester's criterion
hessian_pd[idx] = (
xx < 0 &&
xx*yy-xy*xy > 0 &&
xx*yy*zz + 2*xy*yz*xz - xx*yz*yz - yy*xz*xz - zz*xy*xy < 0
) ? 1 : 0;

}