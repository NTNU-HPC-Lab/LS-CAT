#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradientColumnsKernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
__shared__ float s_Data[COLUMNS_GRAD_BLOCKDIM_Z][COLUMNS_GRAD_BLOCKDIM_X][(COLUMNS_GRAD_RESULT_STEPS + 2 * COLUMNS_GRAD_HALO_STEPS) * COLUMNS_GRAD_BLOCKDIM_Y + 1];

//Offset to the upper halo edge
const int baseX = blockIdx.x * COLUMNS_GRAD_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * COLUMNS_GRAD_RESULT_STEPS - COLUMNS_GRAD_HALO_STEPS) * COLUMNS_GRAD_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * COLUMNS_GRAD_BLOCKDIM_Z + threadIdx.z;
d_Src += (baseZ * imageH + baseY) * imageW + baseX;
d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

//Main data
#pragma unroll

for (int i = COLUMNS_GRAD_HALO_STEPS; i < COLUMNS_GRAD_HALO_STEPS + COLUMNS_GRAD_RESULT_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.x][threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y] = d_Src[i * COLUMNS_GRAD_BLOCKDIM_Y * imageW];
}

//Upper halo
#pragma unroll

for (int i = 0; i < COLUMNS_GRAD_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.x][threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y] = (baseY + i * COLUMNS_GRAD_BLOCKDIM_Y >= 0) ? d_Src[i * COLUMNS_GRAD_BLOCKDIM_Y * imageW] : 0;
}

//Lower halo
#pragma unroll

for (int i = COLUMNS_GRAD_HALO_STEPS + COLUMNS_GRAD_RESULT_STEPS; i < COLUMNS_GRAD_HALO_STEPS + COLUMNS_GRAD_RESULT_STEPS + COLUMNS_GRAD_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.x][threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y]= (baseY + i * COLUMNS_GRAD_BLOCKDIM_Y < imageH) ? d_Src[i * COLUMNS_GRAD_BLOCKDIM_Y * imageW] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll

for (int i = COLUMNS_GRAD_HALO_STEPS; i < COLUMNS_GRAD_HALO_STEPS + COLUMNS_GRAD_RESULT_STEPS; i++) {
float sum = 0;
sum += s_Data[threadIdx.z][threadIdx.x][threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y + 1];
sum -= s_Data[threadIdx.z][threadIdx.x][threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y - 1];
sum *= 0.5f;

d_Dst[i * COLUMNS_GRAD_BLOCKDIM_Y * imageW] = sum;
}
}