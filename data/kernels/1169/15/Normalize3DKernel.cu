#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Normalize3DKernel ( const unsigned short *d_src, const float *d_erosion, const float *d_dilation, float *d_dst, float min_intensity, const int width, const int height, const int depth ) {
const int baseX = blockIdx.x * blockDim.x + threadIdx.x;
const int baseY = blockIdx.y * blockDim.y + threadIdx.y;
const int baseZ = blockIdx.z * blockDim.z + threadIdx.z;

const int idx = (baseZ * height + baseY) * width + baseX;
const float intensity = (float)d_src[idx];
d_dst[idx] = (intensity >= min_intensity) ? (intensity-d_erosion[idx]) / (d_dilation[idx] - d_erosion[idx]) : 0;
}