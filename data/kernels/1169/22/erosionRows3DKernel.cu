#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void erosionRows3DKernel ( unsigned short *d_dst, unsigned short *d_src, int w, int h, int d, int kernel_radius )
{
__shared__ unsigned short smem[ER_ROWS_BLOCKDIM_Z][ER_ROWS_BLOCKDIM_Y][(ER_ROWS_RESULT_STEPS + 2 * ER_ROWS_HALO_STEPS) * ER_ROWS_BLOCKDIM_X];
unsigned short *smem_thread = smem[threadIdx.z][threadIdx.y];

//Offset to the left halo edge
const int baseX = (blockIdx.x * ER_ROWS_RESULT_STEPS - ER_ROWS_HALO_STEPS) * ER_ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ER_ROWS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * ER_ROWS_BLOCKDIM_Z + threadIdx.z;

d_src += (baseZ * h + baseY) * w + baseX;
d_dst += (baseZ * h + baseY) * w + baseX;

//Load main data
#pragma unroll
for (int i = ER_ROWS_HALO_STEPS; i < ER_ROWS_HALO_STEPS + ER_ROWS_RESULT_STEPS; i++) {
smem_thread[threadIdx.x + i * ER_ROWS_BLOCKDIM_X] = d_src[i * ER_ROWS_BLOCKDIM_X];
}

//Load left halo
#pragma unroll
for (int i = 0; i < ER_ROWS_HALO_STEPS; i++) {
smem_thread[threadIdx.x + i * ER_ROWS_BLOCKDIM_X] = (baseX + i * ER_ROWS_BLOCKDIM_X >= 0) ? d_src[i * ER_ROWS_BLOCKDIM_X] : 0;
}

//Load right halo
#pragma unroll
for (int i = ER_ROWS_HALO_STEPS + ER_ROWS_RESULT_STEPS; i < ER_ROWS_HALO_STEPS + ER_ROWS_RESULT_STEPS + ER_ROWS_HALO_STEPS; i++) {
smem_thread[threadIdx.x + i * ER_ROWS_BLOCKDIM_X] = (baseX + i * ER_ROWS_BLOCKDIM_X < w) ? d_src[i * ER_ROWS_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = ER_ROWS_HALO_STEPS; i < ER_ROWS_HALO_STEPS + ER_ROWS_RESULT_STEPS; i++) {
unsigned short *smem_kern = &smem_thread[threadIdx.x + i * ER_ROWS_BLOCKDIM_X - kernel_radius];
unsigned short val = smem_kern[0];

//#pragma unroll
for (int j = 1; j <= 2*kernel_radius; j++) {
val = min(val, smem_kern[j]);
}
d_dst[i * ER_ROWS_BLOCKDIM_X] = val;
}
}