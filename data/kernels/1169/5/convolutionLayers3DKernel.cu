#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionLayers3DKernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int kernel_index, int kernel_radius )
{
__shared__ float s_Data[LAYERS_BLOCKDIM_X][LAYERS_BLOCKDIM_Y][(LAYERS_RESULT_STEPS + 2 * LAYERS_HALO_STEPS) * LAYERS_BLOCKDIM_Z + 1];

//Offset to the upper halo edge
const int baseX = blockIdx.x * LAYERS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * LAYERS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = (blockIdx.z * LAYERS_RESULT_STEPS - LAYERS_HALO_STEPS) * LAYERS_BLOCKDIM_Z + threadIdx.z;
d_Src += (baseZ * imageH + baseY) * imageW + baseX;
d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

const int pitch = imageW*imageH;
const float* kernel = &c_Kernel[kernel_index*MAX_KERNEL_LENGTH];

//Main data
#pragma unroll

for (int i = LAYERS_HALO_STEPS; i < LAYERS_HALO_STEPS + LAYERS_RESULT_STEPS; i++) {
s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_BLOCKDIM_Z] = d_Src[i * LAYERS_BLOCKDIM_Z * pitch];
}

//Upper halo
#pragma unroll

for (int i = 0; i < LAYERS_HALO_STEPS; i++) {
s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_BLOCKDIM_Z] = (baseZ + i * LAYERS_BLOCKDIM_Z >= 0) ? d_Src[i * LAYERS_BLOCKDIM_Z * pitch] : 0;
}

//Lower halo
#pragma unroll

for (int i = LAYERS_HALO_STEPS + LAYERS_RESULT_STEPS; i < LAYERS_HALO_STEPS + LAYERS_RESULT_STEPS + LAYERS_HALO_STEPS; i++) {
s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_BLOCKDIM_Z]= (baseZ + i * LAYERS_BLOCKDIM_Z < imageD) ? d_Src[i * LAYERS_BLOCKDIM_Z * pitch] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll

for (int i = LAYERS_HALO_STEPS; i < LAYERS_HALO_STEPS + LAYERS_RESULT_STEPS; i++) {
float sum = 0;
//#pragma unroll

for (int j = -kernel_radius; j <= kernel_radius; j++) {
sum += kernel[kernel_radius - j] * s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_BLOCKDIM_Z + j];
}

d_Dst[i * LAYERS_BLOCKDIM_Z * pitch] = sum;
}
}