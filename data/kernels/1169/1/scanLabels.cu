#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CCL3D.cu
*/


#define CCL_BLOCK_SIZE_X 8
#define CCL_BLOCK_SIZE_Y 8
#define CCL_BLOCK_SIZE_Z 8

__device__ int d_isNotDone;




__global__ void scanLabels(int* labels, int w, int h, int d) {
const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
const int index = (z*h + y)*w + x;

if (x >= w || y >= h || z >= d) return;

const int Z1 = w*h; const int Y1 = w;

int lcur = labels[index];
if (lcur) {
int lmin = index; // MAX
// 26-neighbors
int lne, pos;
for (int Zdif = -Z1; Zdif <= Z1; Zdif += Z1) {
for (int Ydif = -Y1; Ydif <= Y1; Ydif += Y1) {
for (int Xdif = -1; Xdif <= 1; Xdif += 1) {
pos = index + Zdif + Ydif + Xdif;
lne = (pos >= 0 && pos < w*h*d) ? labels[pos] : 0; // circular boundary
if (lne && lne < lmin) lmin = lne;
}
}
}
// need not (Xdif,Ydif,Zdif)=(0,0,0) but no problem

if (lmin < lcur) {
int lpa = labels[lcur];
labels[lpa] = min(lpa, lmin);
d_isNotDone = 1;
}
}
}