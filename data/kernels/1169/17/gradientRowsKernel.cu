#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradientRowsKernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
__shared__ float s_Data[ROWS_GRAD_BLOCKDIM_Z][ROWS_GRAD_BLOCKDIM_Y][(ROWS_GRAD_RESULT_STEPS + 2 * ROWS_GRAD_HALO_STEPS) * ROWS_GRAD_BLOCKDIM_X];

//Offset to the left halo edge
const int baseX = (blockIdx.x * ROWS_GRAD_RESULT_STEPS - ROWS_GRAD_HALO_STEPS) * ROWS_GRAD_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_GRAD_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * ROWS_GRAD_BLOCKDIM_Z + threadIdx.z;

d_Src += (baseZ * imageH + baseY) * imageW + baseX;
d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

//Load main data
#pragma unroll

for (int i = ROWS_GRAD_HALO_STEPS; i < ROWS_GRAD_HALO_STEPS + ROWS_GRAD_RESULT_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X] = d_Src[i * ROWS_GRAD_BLOCKDIM_X];
}

//Load left halo
#pragma unroll

for (int i = 0; i < ROWS_GRAD_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X] = (baseX + i * ROWS_GRAD_BLOCKDIM_X >= 0) ? d_Src[i * ROWS_GRAD_BLOCKDIM_X] : 0;
}

//Load right halo
#pragma unroll

for (int i = ROWS_GRAD_HALO_STEPS + ROWS_GRAD_RESULT_STEPS; i < ROWS_GRAD_HALO_STEPS + ROWS_GRAD_RESULT_STEPS + ROWS_GRAD_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X] = (baseX + i * ROWS_GRAD_BLOCKDIM_X < imageW) ? d_Src[i * ROWS_GRAD_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll

for (int i = ROWS_GRAD_HALO_STEPS; i < ROWS_GRAD_HALO_STEPS + ROWS_GRAD_RESULT_STEPS; i++)
{
float sum = 0;
sum += s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X + 1];
sum -= s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X - 1];
sum *= 0.5f;

d_Dst[i * ROWS_GRAD_BLOCKDIM_X] = sum;
}
}