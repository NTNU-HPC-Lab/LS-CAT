#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionRows3DKernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int kernel_index, int kernel_radius )
{
__shared__ float s_Data[ROWS_BLOCKDIM_Z][ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

//Offset to the left halo edge
const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * ROWS_BLOCKDIM_Z + threadIdx.z;

d_Src += (baseZ * imageH + baseY) * imageW + baseX;
d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

const float* kernel = &c_Kernel[kernel_index*MAX_KERNEL_LENGTH];

//Load main data
#pragma unroll

for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
}

//Load left halo
#pragma unroll

for (int i = 0; i < ROWS_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X >= 0) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Load right halo
#pragma unroll

for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++) {
s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X < imageW) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll

for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
{
float sum = 0;

//#pragma unroll

for (int j = -kernel_radius; j <= kernel_radius; j++)
{
sum += kernel[kernel_radius - j] * s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
}

d_Dst[i * ROWS_BLOCKDIM_X] = sum;
}
}