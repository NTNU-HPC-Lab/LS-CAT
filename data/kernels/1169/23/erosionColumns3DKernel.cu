#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void erosionColumns3DKernel( unsigned short *d_dst, unsigned short *d_src, int w,int h,int d, int kernel_radius )
{
__shared__ unsigned short smem[ER_COLUMNS_BLOCKDIM_Z][ER_COLUMNS_BLOCKDIM_X][(ER_COLUMNS_RESULT_STEPS + 2 * ER_COLUMNS_HALO_STEPS) * ER_COLUMNS_BLOCKDIM_Y + 1];
unsigned short *smem_thread = smem[threadIdx.z][threadIdx.x];

//Offset to the upper halo edge
const int baseX = blockIdx.x * ER_COLUMNS_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * ER_COLUMNS_RESULT_STEPS - ER_COLUMNS_HALO_STEPS) * ER_COLUMNS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * ER_COLUMNS_BLOCKDIM_Z + threadIdx.z;
d_src += (baseZ * h + baseY) * w + baseX;
d_dst += (baseZ * h + baseY) * w + baseX;

//Main data
#pragma unroll
for (int i = ER_COLUMNS_HALO_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i++) {
smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y] = d_src[i * ER_COLUMNS_BLOCKDIM_Y * w];
}

//Upper halo
#pragma unroll
for (int i = 0; i < ER_COLUMNS_HALO_STEPS; i++) {
smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y] = (baseY + i * ER_COLUMNS_BLOCKDIM_Y >= 0) ? d_src[i * ER_COLUMNS_BLOCKDIM_Y * w] : 0;
}

//Lower halo
#pragma unroll
for (int i = ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS + ER_COLUMNS_HALO_STEPS; i++) {
smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y]= (baseY + i * ER_COLUMNS_BLOCKDIM_Y < h) ? d_src[i * ER_COLUMNS_BLOCKDIM_Y * w] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = ER_COLUMNS_HALO_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i++) {
unsigned short *smem_kern = &smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y - kernel_radius];
unsigned short val = smem_kern[0];

//#pragma unroll
for (int j = 1; j <= 2 * kernel_radius; j++) {
val = min(val, smem_kern[j]);
}
d_dst[i * ER_COLUMNS_BLOCKDIM_Y * w] = val;
}
}