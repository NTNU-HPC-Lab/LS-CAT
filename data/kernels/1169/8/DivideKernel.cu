#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DivideKernel ( float *d_dst, unsigned short *d_denom ) {
const int idx = blockIdx.x;
d_dst[idx] /= d_denom[idx];
}