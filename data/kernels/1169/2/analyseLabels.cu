#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CCL3D.cu
*/


#define CCL_BLOCK_SIZE_X 8
#define CCL_BLOCK_SIZE_Y 8
#define CCL_BLOCK_SIZE_Z 8

__device__ int d_isNotDone;




__global__ void analyseLabels(int* labels, int w, int h, int d) {
const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
const int index = (z*h + y)*w + x;

if (x >= w || y >= h || z >= d) return;

int lcur = labels[index];
if (lcur) {
int r = labels[lcur];
while(r != lcur) {
lcur = labels[r];
r = labels[lcur];
}
labels[index] = lcur;
}
}