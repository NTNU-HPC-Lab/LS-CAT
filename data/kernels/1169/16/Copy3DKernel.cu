#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Copy3DKernel ( const unsigned short *d_src, float *d_dst, float min_intensity, const int width, const int height, const int depth ) {
const int baseX = blockIdx.x * blockDim.x + threadIdx.x;
const int baseY = blockIdx.y * blockDim.y + threadIdx.y;
const int baseZ = blockIdx.z * blockDim.z + threadIdx.z;

const int idx = (baseZ * height + baseY) * width + baseX;
const float intensity = (float)d_src[idx];
d_dst[idx] = (intensity >= min_intensity) ? intensity : 0;
}