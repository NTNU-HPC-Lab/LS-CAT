#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initGridKernel ( float *d_grid, int axis, int w, int h, int d ) {
const int baseX = blockIdx.x * IG_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * IG_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * IG_BLOCKDIM_Z + threadIdx.z;

const int idx = (baseZ * h + baseY) * w + baseX;

if(axis == 0) {
d_grid[idx] = (float)baseX;
} else if(axis == 1) {
d_grid[idx] = (float)baseY;
} else {
d_grid[idx] = (float)baseZ;
}

}