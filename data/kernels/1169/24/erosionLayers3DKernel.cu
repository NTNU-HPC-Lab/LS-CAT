#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void erosionLayers3DKernel( unsigned short *d_dst, unsigned short *d_src, int w, int h, int d, int kernel_radius )
{
__shared__ unsigned short smem[ER_LAYERS_BLOCKDIM_X][ER_LAYERS_BLOCKDIM_Y][(ER_LAYERS_RESULT_STEPS + 2 * ER_LAYERS_HALO_STEPS) * ER_LAYERS_BLOCKDIM_Z + 1];
unsigned short *smem_thread = smem[threadIdx.x][threadIdx.y];

//Offset to the upper halo edge
const int baseX = blockIdx.x * ER_LAYERS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ER_LAYERS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = (blockIdx.z * ER_LAYERS_RESULT_STEPS - ER_LAYERS_HALO_STEPS) * ER_LAYERS_BLOCKDIM_Z + threadIdx.z;
d_src += (baseZ * h + baseY) * w + baseX;
d_dst += (baseZ * h + baseY) * w + baseX;

const int pitch = w*h;

//Main data
#pragma unroll
for (int i = ER_LAYERS_HALO_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i++) {
smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z] = d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch];
}

//Upper halo
#pragma unroll
for (int i = 0; i < ER_LAYERS_HALO_STEPS; i++) {
smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z] = (baseZ + i * ER_LAYERS_BLOCKDIM_Z >= 0) ? d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch] : 0;
}

//Lower halo
#pragma unroll
for (int i = ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS + ER_LAYERS_HALO_STEPS; i++) {
smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z]= (baseZ + i * ER_LAYERS_BLOCKDIM_Z < d) ? d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = ER_LAYERS_HALO_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i++) {
unsigned short *smem_kern = &smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z - kernel_radius];
unsigned short val = smem_kern[0];

//#pragma unroll
for (int j = 1; j <= 2*kernel_radius; j++) {
val = min(val, smem_kern[j]);
}
d_dst[i * ER_LAYERS_BLOCKDIM_Z * pitch] = val;
}
}