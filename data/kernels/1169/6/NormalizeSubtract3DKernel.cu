#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeSubtract3DKernel(float * img_src, const float * img_sub, const int width, const int height, const int depth, float normalizer) {
const int baseX = blockIdx.x * SUB_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * SUB_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z * SUB_BLOCKDIM_Z + threadIdx.z;

const int idx = (baseZ * height + baseY) * width + baseX;
img_src[idx] = (img_src[idx] - img_sub[idx]) * normalizer;

}