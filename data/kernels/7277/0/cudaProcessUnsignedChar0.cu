#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaProcessUnsignedChar0(unsigned char *dst, unsigned char *src, int imgW, int imgH)
{
int tx = threadIdx.x;
int ty = threadIdx.y;
int bw = blockDim.x;
int bh = blockDim.y;
int x = blockIdx.x*bw + tx * 2;
int y = blockIdx.y*bh + ty * 2;
int px = y * imgW + x;

bool flag = 0 < y && y < (imgH - 2) && 0 < x && x < (imgW - 2);
int sx1 = flag ? px - imgW : 0;
int sx2 = flag ? px - imgW + 1 : 0;
int sx3 = flag ? px - imgW + 2 : 0;
int sx4 = flag ? px - 1 : 0;
int sx5 = flag ? px : 0;
int sx6 = flag ? px + 1 : 0;
int sx7 = flag ? px + 2 : 0;
int sx8 = flag ? px + imgW - 1 : 0;
int sx9 = flag ? px + imgW : 0;
int sxa = flag ? px + imgW + 1 : 0;
int sxb = flag ? px + imgW + 2 : 0;
int sxc = flag ? px + imgW * 2 - 1 : 0;
int sxd = flag ? px + imgW * 2     : 0;
int sxe = flag ? px + imgW * 2 + 1 : 0;

// G0 R0 G1 R1    x0 x1 x2 x3
// B0 G2 B1 G3    x4 x5 x6 x7
// G4 R2 G5 R3    x8 x9 xA xB
// B2 G6 B3 G7    xC xD xE xF

int g1 = (int)src[sx2];
int g2 = (int)src[sx5];
int g3 = (int)src[sx7];
int g4 = (int)src[sx8];
int g5 = (int)src[sxa];
int g6 = (int)src[sxd];
int b0 = (int)src[sx4];
int b1 = (int)src[sx6];
int b2 = (int)src[sxc];
int b3 = (int)src[sxe];
int r0 = (int)src[sx1];
int r1 = (int)src[sx3];
int r2 = (int)src[sx9];
int r3 = (int)src[sxb];

int db0 = (b0 + b1) >> 1;
int dg0 = g2;
int dr0 = (r0 + r1) >> 1;
int db1 = b1;
int dg1 = (g1 + g2 + g3 + g5) >> 2;
int dr1 = (r0 + r1 + r2 + r3) >> 2;
int db2 = (b0 + b1 + b2 + b3) >> 2;
int dg2 = (g2 + g4 + g5 + g6) >> 2;
int dr2 = r2;
int db3 = (b1 + b3) >> 1;
int dg3 = g5;
int dr3 = (r2 + r3) >> 1;

int dx = px * 3;
int dst0 = dx;
int dst1 = dx + 3;
int dst2 = dx + imgW * 3;
int dst3 = dx + (imgW + 1) * 3;
dst[dst0 + 0 < imgW * imgH * 3 ? dst0 + 0 : 0] = (unsigned char)db0;
dst[dst0 + 1 < imgW * imgH * 3 ? dst0 + 1 : 0] = (unsigned char)dg0;
dst[dst0 + 2 < imgW * imgH * 3 ? dst0 + 2 : 0] = (unsigned char)dr0;
dst[dst1 + 0 < imgW * imgH * 3 ? dst1 + 0 : 0] = (unsigned char)db1;
dst[dst1 + 1 < imgW * imgH * 3 ? dst1 + 1 : 0] = (unsigned char)dg1;
dst[dst1 + 2 < imgW * imgH * 3 ? dst1 + 2 : 0] = (unsigned char)dr1;
dst[dst2 + 0 < imgW * imgH * 3 ? dst2 + 0 : 0] = (unsigned char)db2;
dst[dst2 + 1 < imgW * imgH * 3 ? dst2 + 1 : 0] = (unsigned char)dg2;
dst[dst2 + 2 < imgW * imgH * 3 ? dst2 + 2 : 0] = (unsigned char)dr2;
dst[dst3 + 0 < imgW * imgH * 3 ? dst3 + 0 : 0] = (unsigned char)db3;
dst[dst3 + 1 < imgW * imgH * 3 ? dst3 + 1 : 0] = (unsigned char)dg3;
dst[dst3 + 2 < imgW * imgH * 3 ? dst3 + 2 : 0] = (unsigned char)dr3;
}