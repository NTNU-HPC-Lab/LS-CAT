#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_product(float *a, float *b, float *c)
{
c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
}