#include "hip/hip_runtime.h"
#include "includes.h"
#define PI 3.141592653589793
#define BLOCKSIZE 1024


__global__ void cuAdd(float *dst, float *src, int size)
{
int id=blockIdx.x*blockDim.x+threadIdx.x;
if(id>=size) return;
dst[id]+=src[id];
}