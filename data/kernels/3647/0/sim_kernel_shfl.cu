#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__device__ inline double warp_accel_shfl(double z, double d_2inv, int shfl_mask) {
return d_2inv*(__shfl_down_sync(shfl_mask, z, 1) + __shfl_up_sync(shfl_mask, z, 1) - 2.0*z);
}
__global__ void sim_kernel_shfl(double *z, double *v, size_t nx, size_t ny, double dx2inv, double dy2inv, double dt) {
__shared__ double z_tile[WARP_SIZE][WARP_SIZE];
__shared__ double ay_tile[WARP_SIZE][WARP_SIZE];

const int block_mesh_x = warpSize*blockIdx.x + 1;
const int block_mesh_y = warpSize*blockIdx.y + 1;

const int mesh_xx = block_mesh_x + threadIdx.x;
const int mesh_xy = block_mesh_y + threadIdx.y;
const int mesh_yx = block_mesh_x + threadIdx.y;
const int mesh_yy = block_mesh_y + threadIdx.x;

const double z_val_x = z_tile[threadIdx.y][threadIdx.x] = IDX2D(z, mesh_xy, nx, mesh_xx);

if (mesh_xx >= nx-1 || mesh_xy >= ny-1 /*|| mesh_yx > nx-1 || mesh_yy >= ny-1*/)
return;

__syncthreads();

const double z_val_y = z_tile[threadIdx.x][threadIdx.y];

//    const int shfl_mask = 0x7 << (threadIdx.x - 1);
const int shfl_mask = 0x7 << (threadIdx.x - 1);

double ax = warp_accel_shfl(z_val_x, dx2inv, shfl_mask);
double ay = warp_accel_shfl(z_val_y, dy2inv, shfl_mask);
if (threadIdx.x == 0 || threadIdx.x == warpSize-1) {
const int n = threadIdx.x == 0 ? -1 : +1;
ax = dx2inv*(IDX2D(z, mesh_xy, nx, mesh_xx+n) + z_tile[threadIdx.y][threadIdx.x-n]
- 2.0*z_val_x);
ay = dy2inv*(IDX2D(z, mesh_yy+n, nx, mesh_yx) + z_tile[threadIdx.x-n][threadIdx.y]
- 2.0*z_val_y);
}

ay_tile[threadIdx.x][threadIdx.x] = ay;
__syncthreads();
ay = ay_tile[threadIdx.y][threadIdx.x];

const double v_val = (IDX2D(v, mesh_xy, nx, mesh_xx) += (ax+ay)/2.0*dt);
IDX2D(z, mesh_xy, nx, mesh_xx) += dt*v_val;
}