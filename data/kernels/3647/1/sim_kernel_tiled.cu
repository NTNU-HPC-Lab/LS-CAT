#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void sim_kernel_tiled(double *z, double *v, size_t nx, size_t ny, double dx2inv, double dy2inv, double dt) {
extern __shared__ double z_tile[];

const int block_mesh_x = blockDim.x*blockIdx.x + 1;
const int block_mesh_y = blockDim.y*blockIdx.y + 1;

const int mesh_xx = block_mesh_x + threadIdx.x;
const int mesh_xy = block_mesh_y + threadIdx.y;

// We have to read into the tile BEFORE dropping threads so that it's actually fully
// initialized!
const double z_val = IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x)
= IDX2D(z, mesh_xy, nx, mesh_xx);

if (mesh_xx >= nx-1 || mesh_xy >= ny-1)
return;

__syncthreads();

double ax, ay;
if (1 <= threadIdx.x && threadIdx.x <= blockDim.x-2)
ax = dx2inv*(IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x-1)
+ IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x+1)
- 2.0*z_val);
else {
const int n = threadIdx.x == 0 ? -1 : +1;
ax = dx2inv*(IDX2D(z, mesh_xy, nx, mesh_xx+n)
+ IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x-n)
- 2.0*z_val);
}

if (1 <= threadIdx.y && threadIdx.y <= blockDim.y-2)
ay = dy2inv*(IDX2D(z_tile, threadIdx.y-1, blockDim.x, threadIdx.x)
+ IDX2D(z_tile, threadIdx.y+1, blockDim.x, threadIdx.x)
- 2.0*z_val);
else {
const int n = threadIdx.y == 0 ? -1 : +1;
ay = dx2inv*(IDX2D(z, mesh_xy+n, nx, mesh_xx)
+ IDX2D(z_tile, threadIdx.y-n, blockDim.x, threadIdx.x)
- 2.0*z_val);
}

const double v_val = IDX2D(v, mesh_xy, nx, mesh_xx) += (ax+ay)/2.0*dt;
IDX2D(z, mesh_xy, nx, mesh_xx) += dt*v_val;
}