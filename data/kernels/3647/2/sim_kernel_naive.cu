#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void sim_kernel_naive(double *z, double *v, size_t nx, size_t ny, double dx2inv, double dy2inv, double dt) {
const int mesh_x = blockIdx.x*blockDim.x + threadIdx.x + 1;
const int mesh_y = blockIdx.y*blockDim.y + threadIdx.y + 1;

if (mesh_x >= nx-1 || mesh_y >= ny-1) return;

const double z_val = IDX2D(z, mesh_y, nx, mesh_x);
const double ax = dx2inv*(IDX2D(z, mesh_y, nx, mesh_x-1) + IDX2D(z, mesh_y, nx, mesh_x+1)
- 2.0*z_val);
const double ay = dy2inv*(IDX2D(z, mesh_y-1, nx, mesh_x) + IDX2D(z, mesh_y+1, nx, mesh_x)
- 2.0*z_val);

double const v_val = IDX2D(v, mesh_y, nx, mesh_x) + dt*(ax + ay)/2;
IDX2D(v, mesh_y, nx, mesh_x) = v_val;
IDX2D(z, mesh_y, nx, mesh_x) += dt*v_val;
}