#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void grayscale_kernel(double *z, unsigned char *output, size_t size, double z_min, double z_max) {
const double grid_size = blockDim.x*gridDim.x;
const int idx = threadIdx.x + blockDim.x*blockIdx.x;

for (int i = idx; i < size; i += grid_size)
output[i] = (char) round((z[i]-z_min)/(z_max-z_min)*255);
}