#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void buildCompactedIndexKernel( const unsigned* valid_indicator, const unsigned table_size, unsigned* compacted_index ) {
const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
if(idx < table_size) {
unsigned offset = 0xffffffffu;
if(valid_indicator[idx] > 0) {
offset = compacted_index[idx] - 1;
}
compacted_index[idx] = offset;
}
}