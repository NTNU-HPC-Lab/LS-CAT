#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compactIndicatorToPixelKernel( const unsigned* candidate_pixel_indicator, const unsigned* prefixsum_indicator, unsigned img_cols, ushort2* compacted_pixels ) {
const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
if(candidate_pixel_indicator[idx] > 0) {
const auto offset = prefixsum_indicator[idx] - 1;
const unsigned short x = idx % img_cols;
const unsigned short y = idx / img_cols;
compacted_pixels[offset] = make_ushort2(x, y);
}
}