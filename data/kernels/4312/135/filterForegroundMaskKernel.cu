#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void filterForegroundMaskKernel( hipTextureObject_t foreground_mask, unsigned mask_rows, unsigned mask_cols, const float sigma, hipSurfaceObject_t filter_mask ) {
const int x = threadIdx.x + blockDim.x * blockIdx.x;
const int y = threadIdx.y + blockDim.y * blockIdx.y;
if(x >= mask_cols || y >= mask_rows) return;

//A window search
const int halfsize = __float2uint_ru(sigma) * 2;
float total_weight = 0.0f;
float total_value = 0.0f;
for(int neighbor_y = y - halfsize; neighbor_y <= y + halfsize; neighbor_y++) {
for(int neighbor_x = x - halfsize; neighbor_x <= x + halfsize; neighbor_x++) {
//Retrieve the mask value at neigbour
const unsigned char neighbor_foreground = tex2D<unsigned char>(foreground_mask, neighbor_x, neighbor_y);

//Compute the gaussian weight
const float diff_x_square = (neighbor_x - x) * (neighbor_x - x);
const float diff_y_square = (neighbor_y - y) * (neighbor_y - y);
const float weight = __expf(0.5f * (diff_x_square + diff_y_square) / (sigma * sigma));

//Accumlate it
if(neighbor_x >= 0 && neighbor_x < mask_cols && neighbor_y >= 0 && neighbor_y < mask_rows)
{
total_weight += weight;
total_value += weight * float(1 - neighbor_foreground);
}
}
}


//Compute the value locally
const unsigned char foreground_indicator = tex2D<unsigned char>(foreground_mask, x, y);
float filter_value = 0.0;
if(foreground_indicator == 0) {
filter_value = total_value / (total_weight + 1e-3f);
}


//Write to the surface
surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
}