#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void markPotentialMatchedDepthPairKernel( hipTextureObject_t index_map, unsigned img_rows, unsigned img_cols, unsigned* reference_pixel_matched_indicator ) {
const auto x = threadIdx.x + blockDim.x*blockIdx.x;
const auto y = threadIdx.y + blockDim.y*blockIdx.y;
if (x >= img_cols || y >= img_rows) return;

//The indicator will must be written to pixel_occupied_array
const auto offset = y * img_cols + x;

//Read the value on index map
const auto surfel_index = tex2D<unsigned>(index_map, x, y);

//Need other criterion?
unsigned indicator = 0;
if(surfel_index != d_invalid_index) {
indicator = 1;
}

reference_pixel_matched_indicator[offset] = indicator;
}