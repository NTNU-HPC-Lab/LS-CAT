#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assembleBinBlockCSRKernel( const unsigned matrix_size, const float* diagonal_blks, const float* nondiagonal_blks, const int* csr_rowptr, const unsigned* blkrow_offset, float* JtJ_data ) {
const auto row_idx = threadIdx.x + blockDim.x * blockIdx.x;
if(row_idx >= matrix_size) return;

//Now the query should all be safe
int data_offset = csr_rowptr[row_idx];
const auto blkrow_idx = row_idx / 6;
const auto inblk_offset = row_idx % 6;

//First fill the diagonal blks
for (auto k = 0; k < 6; k++, data_offset += bin_size) {
JtJ_data[data_offset] = diagonal_blks[36 * blkrow_idx + inblk_offset + 6 * k];
}

//Next fill the non-diagonal blks
auto Iij_begin = blkrow_offset[blkrow_idx];
const auto Iij_end = blkrow_offset[blkrow_idx + 1];
for (; Iij_begin < Iij_end; Iij_begin++) {
for (int k = 0; k < 6; k++, data_offset += bin_size) {
JtJ_data[data_offset] = nondiagonal_blks[36 * Iij_begin + inblk_offset + 6 * k];
}
}
}