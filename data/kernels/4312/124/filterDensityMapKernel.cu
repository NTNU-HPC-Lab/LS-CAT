#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void filterDensityMapKernel( hipTextureObject_t density_map, unsigned rows, unsigned cols, hipSurfaceObject_t filter_density_map ) {
const auto x = threadIdx.x + blockIdx.x * blockDim.x;
const auto y = threadIdx.y + blockIdx.y * blockDim.y;
if(x >= cols || y >= rows) return;

const auto half_width = 5;
const float center_density = tex2D<float>(density_map, x, y);

//The window search
float sum_all = 0.0f; float sum_weight = 0.0f;
for(auto y_idx = y - half_width; y_idx <= y + half_width; y_idx++) {
for(auto x_idx = x - half_width; x_idx <= x + half_width; x_idx++) {
const float density = tex2D<float>(density_map, x_idx, y_idx);
const float value_diff2 = (center_density - density) * (center_density - density);
const float pixel_diff2 = (x_idx - x) * (x_idx - x) + (y_idx - y) * (y_idx - y);
const float this_weight = (density > 0.0f) * expf(-(1.0f / 25) * pixel_diff2) * expf(-(1.0f / 0.01) * value_diff2);
sum_weight += this_weight;
sum_all += this_weight * density;
}
}

//The filter value
float filter_density_value = sum_all / (sum_weight);

//Clip the value to suitable range
if(filter_density_value >= 1.0f) {
filter_density_value = 1.0f;
} else if(filter_density_value >= 0.0f) {
//pass
} else {
filter_density_value = 0.0f;
}
//if(isnan(filter_density_value)) printf("Nan in the image");
surf2Dwrite(filter_density_value, filter_density_map, x * sizeof(float), y);
}