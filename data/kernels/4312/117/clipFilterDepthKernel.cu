#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipFilterDepthKernel( hipTextureObject_t raw_depth, const unsigned clip_img_rows, const unsigned clip_img_cols, const unsigned clip_near, const unsigned clip_far, const float sigma_s_inv_square, const float sigma_r_inv_square, hipSurfaceObject_t filter_depth ) {
//Parallel over the clipped image
const auto x = threadIdx.x + blockDim.x * blockIdx.x;
const auto y = threadIdx.y + blockDim.y * blockIdx.y;
if (y >= clip_img_rows || x >= clip_img_cols) return;

//Compute the center on raw depth
const auto half_width = 5;
const auto raw_x = x + boundary_clip;
const auto raw_y = y + boundary_clip;
const unsigned short center_depth = tex2D<unsigned short>(raw_depth, raw_x, raw_y);

//Iterate over the window
float sum_all = 0.0f; float sum_weight = 0.0f;
for(auto y_idx = raw_y - half_width; y_idx <= raw_y + half_width; y_idx++) {
for(auto x_idx = raw_x - half_width; x_idx <= raw_x + half_width; x_idx++) {
const unsigned short depth = tex2D<unsigned short>(raw_depth, x_idx, y_idx);
const float depth_diff2 = (depth - center_depth) * (depth - center_depth);
const float pixel_diff2 = (x_idx - raw_x) * (x_idx - raw_x) + (y_idx - raw_y) * (y_idx - raw_y);
const float this_weight = (depth > 0) * expf(-sigma_s_inv_square * pixel_diff2) * expf(-sigma_r_inv_square * depth_diff2);
sum_weight += this_weight;
sum_all += this_weight * depth;
}
}

//Put back to the filtered depth
unsigned short filtered_depth_value = __float2uint_rn(sum_all / sum_weight);
if (filtered_depth_value < clip_near || filtered_depth_value > clip_far) filtered_depth_value = 0;
surf2Dwrite(filtered_depth_value, filter_depth, x * sizeof(unsigned short), y);
}