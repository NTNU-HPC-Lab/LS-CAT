#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void markValidIndexMapPixelKernel( hipTextureObject_t index_map, int validity_halfsize, unsigned img_rows, unsigned img_cols, unsigned char* flatten_validity_indicator ) {
const auto x_center = threadIdx.x + blockDim.x * blockIdx.x;
const auto y_center = threadIdx.y + blockDim.y * blockIdx.y;
if(x_center >= img_cols || y_center >= img_rows) return;
const auto offset = x_center + y_center * img_cols;

//Only depend on this pixel
if(validity_halfsize <= 0) {
const auto surfel_index = tex2D<unsigned>(index_map, x_center, y_center);
unsigned char validity = 0;
if(surfel_index != 0xFFFFFFFF) validity = 1;

//Write it and return
flatten_validity_indicator[offset] = validity;
return;
}

//Should perform a window search as the halfsize is at least 1
unsigned char validity = 1;
for(auto y = y_center - validity_halfsize; y <= y_center + validity_halfsize; y++) {
for(auto x = x_center - validity_halfsize; x <= x_center + validity_halfsize; x++) {
if(tex2D<unsigned>(index_map, x, y) == 0xFFFFFFFF) validity = 0;
}
}

//Save it
flatten_validity_indicator[offset] = validity;
}