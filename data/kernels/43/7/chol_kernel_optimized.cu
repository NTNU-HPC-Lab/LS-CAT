#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized(float * U, int k, int stride) {
//With stride...

//Iterators
unsigned int j;
unsigned int num_rows = MATRIX_SIZE;


//This call acts as a single K iteration
//Each block does a single i iteration
//Need to consider offset,
int i = blockIdx.x + (k + 1);
//Each thread does some part of j
//Stide in units of 'stride'
//Thread 0 does 0, 16, 32
//Thread 1 does 1, 17, 33
//..etc.
int offset = i; //From original loop
int jstart = threadIdx.x + offset;
int jstep = stride;

//Only continue if in bounds?
//Top limit on i for whole (original) loop
int jtop = num_rows - 1;
//Bottom limit on i for whole (original) loop
int jbottom = i;

//Do work for this i iteration
//Want to stride across
for (j = jstart; (j >= jbottom) && (j <= jtop); j += jstep) {
U[i * num_rows + j] -= U[k * num_rows + i] * U[k * num_rows + j];
}
}