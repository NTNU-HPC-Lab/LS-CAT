#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_cudaUFMG_zero(float * U, int elem_per_thr) {
// Get a thread identifier
int tx = blockIdx.x * blockDim.x + threadIdx.x;
int ty = blockIdx.y * blockDim.y + threadIdx.y;

int tn = ty * blockDim.x * gridDim.x + tx;

for(unsigned i=0;i<elem_per_thr;i++){
int iel = tn * elem_per_thr + i;
int xval = iel % MATRIX_SIZE;
int yval = iel / MATRIX_SIZE;

if(xval == yval){
continue;
}

// if on the upper diagonal...
if(yval < xval){
xval = MATRIX_SIZE - xval - 1;
yval = MATRIX_SIZE - yval - 1;
}
int iU = xval + yval * MATRIX_SIZE;
U[iU] = 0;
}

}