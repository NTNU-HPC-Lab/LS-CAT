#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized_div_old(float * U, int k, int stride) {
//General thread id
int tx = blockIdx.x * blockDim.x + threadIdx.x;

//Iterators
unsigned int j;
unsigned int num_rows = MATRIX_SIZE;

//Only let one thread do this
if (tx == 0) {
// Take the square root of the diagonal element
U[k * num_rows + k] = sqrt(U[k * num_rows + k]);
//Don't bother doing check...live life on the edge!

// Division step
for (j = (k + 1); j < num_rows; j++) {
U[k * num_rows + j] /= U[k * num_rows + k]; // Division step
}
}
}