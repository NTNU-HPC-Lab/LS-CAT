#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized_div(float * U, int k, int stride) {
//With stride...

//General thread id
int tx = blockIdx.x * blockDim.x + threadIdx.x;

//Iterators
unsigned int j;
unsigned int num_rows = MATRIX_SIZE;

//Only let one thread do this
if (tx == 0) {
// Take the square root of the diagonal element
U[k * num_rows + k] = sqrt(U[k * num_rows + k]);
//Don't bother doing check...live life on the edge!
}

//Each thread does some part of j
//Stide in units of 'stride'
//Thread 0 does 0, 16, 32
//Thread 1 does 1, 17, 33
//..etc.
int offset = (k + 1); //From original loop
int jstart = threadIdx.x + offset;
int jstep = stride;

//Only continue if in bounds?
//Top limit on i for whole (original) loop
int jtop = num_rows - 1;
//Bottom limit on i for whole (original) loop
int jbottom = (k + 1);

//Do work for this i iteration
//Division step
//Only let one thread block do this
if (blockIdx.x == 0) {
for (j = jstart; (j >= jbottom) && (j <= jtop); j += jstep) {
U[k * num_rows + j] /= U[k * num_rows + k]; // Division step
}
}
}