#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized_no_stride(float * U, int k, int stride) {
//Iterators
unsigned int j;
unsigned int num_rows = MATRIX_SIZE;

//TODO USE STRIDE

//This call acts as a single K iteration
//Each block does a single i iteration
//Need to consider offset,
int i = blockIdx.x + (k + 1);
//Each thread does some part of j
//Split j based on stride and thread index
//Index 0 is j= 0-15
//Index 1 is j=16-31
int offset = i;
int jstart = (threadIdx.x * stride) + offset;
int jend = jstart + (stride - 1);

//Only continue if in bounds?
//Top limit on i for whole (original) loop
int jtop = num_rows - 1;
//Bottom limit on i for whole (original) loop
int jbottom = i;
//Check boundaries, else do nothing
if (!((jstart >= jbottom) && (jend <= jtop))) {
return; //This thread does nothing now
}

//Do work  for this i iteration
//Want to stride across
for (j = jstart; j <= jend; j++) {
U[i * num_rows + j] -= U[k * num_rows + i] * U[k * num_rows + j];
}
}