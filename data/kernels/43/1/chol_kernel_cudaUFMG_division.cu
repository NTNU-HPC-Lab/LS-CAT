#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_cudaUFMG_division(float * U, int elem_per_thr) {
// Get a thread identifier
int tx = blockIdx.x * blockDim.x + threadIdx.x;
int ty = blockIdx.y * blockDim.y + threadIdx.y;

int tn = ty * blockDim.x * gridDim.x + tx;


//#define DEBUGDIV

#ifdef DEBUGDIV

int dbg = 0;
if(blockIdx.x == 4){
if(blockIdx.y == 5){
if(threadIdx.x == 2){
if(threadIdx.y == 1){
dbg = 1;
printf("\n\n");
printf("\ntx=%d \nty=%d \ntn=%d", tx, ty, tn);
}
}
}
}

#endif


for(unsigned i=0;i<elem_per_thr;i++){
int iel = tn * elem_per_thr + i;
int xval = iel % MATRIX_SIZE;
int yval = iel / MATRIX_SIZE;

if(xval == yval){
continue;
}


#ifdef     DEBUGDIV
if(dbg == 1){
if(i==37){
printf("\niel=%d \nxval=%d \nyval=%d", iel, xval, yval);
}
}
#endif


// if on the lower diagonal...
if(yval > xval){
xval = MATRIX_SIZE - xval - 1;
yval = MATRIX_SIZE - yval - 1;
}


int iU = xval + yval * MATRIX_SIZE;
int iDiag = yval + yval * MATRIX_SIZE;

#ifdef     DEBUGDIV
if(dbg == 1){
if(i==37){
printf("\nxtrans=%d \nytrans=%d \niU=%d \niDiag=%d", xval, yval, iU, iDiag);
printf("\n\n");
}
}
#endif

U[iU] /= U[iDiag];

}

}