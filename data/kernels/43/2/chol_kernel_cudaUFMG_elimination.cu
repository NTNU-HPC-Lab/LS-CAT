#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_cudaUFMG_elimination(float * U, int k) {

//This call acts as a single K iteration
//Each block does a single i iteration
//Need to consider offset,
int i = (k+1) + blockIdx.x;

//Each thread does some part of j
//Stide in units of 'stride'
//Thread 0 does 0, 16, 32
//Thread 1 does 1, 17, 33
//..etc.
int jstart = i + threadIdx.x;
int jstep = blockDim.x;

// Pre-calculate indexes
int kM = k * MATRIX_SIZE;
int iM = i * MATRIX_SIZE;
int ki = kM + i;

//Do work for this i iteration
//Want to stride across
for (int j=jstart; j<MATRIX_SIZE; j+=jstep) {
U[iM + j] -= U[ki] * U[kM + j];
}
}