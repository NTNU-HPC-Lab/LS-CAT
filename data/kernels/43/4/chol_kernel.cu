#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel(float * U, int ops_per_thread) {
//Determine the boundaries for this thread
//Get a thread identifier
int tx = blockIdx.x * blockDim.x + threadIdx.x;

//Iterators
unsigned int i, j, k;
//unsigned int size = MATRIX_SIZE*MATRIX_SIZE;
unsigned int num_rows = MATRIX_SIZE;

//Contents of the A matrix should already be in U

//Perform the Cholesky decomposition in place on the U matrix
for (k = 0; k < num_rows; k++) {
//Only one thread does squre root and division
if (tx == 0) {
// Take the square root of the diagonal element
U[k * num_rows + k] = sqrt(U[k * num_rows + k]);
//Don't bother doing check...live life on the edge!

// Division step
for (j = (k + 1); j < num_rows; j++) {
U[k * num_rows + j] /= U[k * num_rows + k]; // Division step
}
}

//Sync threads!!!!! (only one thread block so, ok)
__syncthreads();

//Elimination step

int istart = ( k + 1 )  +  tx * ops_per_thread;
int iend = istart + ops_per_thread;

for (i = istart; i < iend; i++) {
//Do work  for this i iteration
for (j = i; j < num_rows; j++) {
U[i * num_rows + j] -= U[k * num_rows + i] * U[k * num_rows + j];
}
}


//Sync threads!!!!! (only one thread block so, ok)
__syncthreads();
}

//Sync threads!!!!! (only one thread block so, ok)
__syncthreads();




//As the final step, zero out the lower triangular portion of U
//for(i = 0; i < U.num_rows; i++)
//Each thread does so many iterations of zero out loop
//Starting index for this thread
int istart = tx * ops_per_thread;
//Ending index for this thread
int iend = istart + ops_per_thread;

//Check boundaries, else do nothing
for (i = istart; i < iend; i++) {
//Do work  for this i iteration
for (j = 0; j < i; j++) {
U[i * num_rows + j] = 0.0;
}
}


//Don't sync, will sync outside here
}