#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned int*) end_ptr) {
return;
}

for (i = offset;i < BLOCKSIZE/sizeof(unsigned int); i+=MOD_SZ){
ptr[i] =p1;
}

for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i++){
if (i % MOD_SZ != offset){
ptr[i] =p2;
}
}

return;
}