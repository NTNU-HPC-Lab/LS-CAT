#include "hip/hip_runtime.h"
#include "includes.h"

#define gpu_assert(rv) gpu_assert_h((rv), __FILE__, __LINE__)
__global__ void kernel()
{
printf("Hello Kernel %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}