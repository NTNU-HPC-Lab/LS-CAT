#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bit_reduce_count(const uint32_t *input_array, uint32_t *intBuf, uint32_t *countBuf, const uint16_t dataCount) {
extern __shared__ uint32_t sharedMem[];

const uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
const uint32_t a = input_array[i];
uint8_t size = 0;
if (a <= 0xff) {
size = 1;
} else if (a <= 0xffff) {
size = sizeof(uint16_t);
} else {
size = sizeof(uint32_t);
}

sharedMem[threadIdx.x] = size;

__syncthreads();

// really dumb addition
if (threadIdx.x == 1) {
uint32_t total = 0;
for (uint16_t i = 0; i < dataCount; i++) {
total += sharedMem[i];
sharedMem[i] = total;
}
countBuf[blockIdx.x] = total;
}
__syncthreads();

// block comapct
uint8_t* writeindex = (threadIdx.x + sharedMem[threadIdx.x] - size) + ((uint8_t*)&intBuf[(blockDim.x * blockIdx.x)]);
//uint8_t* writeindex = (threadIdx.x + sharedMem[threadIdx.x] - size) + ((uint8_t*)&intBuf[0]);

if (a <= 0xff) {
*writeindex = 1;
uint8_t b = static_cast<uint8_t>(a);
memcpy(writeindex+1, &b, sizeof(uint8_t));
} else if (a <= 0xffff) {
*writeindex = sizeof(uint16_t);
uint16_t s = static_cast<uint16_t>(a);
memcpy(writeindex+1, &s, sizeof(uint16_t));
} else {
*writeindex = sizeof(uint32_t);
memcpy(writeindex+1, &a, sizeof(uint32_t));
}

}