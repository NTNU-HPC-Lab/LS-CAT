#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void seq_compact(uint8_t *intBuf, const uint16_t dataSize, uint32_t *sizeBuf) {
uint16_t writeIndex = 0;
for (uint16_t i = 0; i < dataSize; ++i) {
const uint16_t readIndex = i * 4;
uint8_t size = intBuf[readIndex];
memcpy(&intBuf[writeIndex], &intBuf[readIndex], size + 1);
writeIndex += size + 1;
}
sizeBuf[0] = writeIndex;

// zero out the rest of the buffer
const uint32_t int_buf_size = (dataSize * sizeof(uint32_t)) + (dataSize * sizeof(uint8_t));
memset(&intBuf[writeIndex], 0, int_buf_size - int_buf_size);
}