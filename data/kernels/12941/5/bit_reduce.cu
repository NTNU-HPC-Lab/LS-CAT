#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bit_reduce(const uint32_t *input_array, uint32_t *intBuf) {
uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t a = input_array[i];

if (a <= 0xff) {
intBuf[i] = 1;
uint8_t b = static_cast<uint8_t>(a);
memcpy((uint8_t *)(&intBuf[i]) + 1, &b, sizeof(uint8_t));
} else if (a <= 0xffff) {
intBuf[i] = sizeof(uint16_t);
uint16_t s = static_cast<uint16_t>(a);
memcpy((uint8_t *)(&intBuf[i]) + 1, &s, sizeof(uint16_t));
} else {
intBuf[i] = sizeof(uint32_t);
memcpy((uint8_t *)(&intBuf[i]) + 1, &a, sizeof(uint32_t));
}
}