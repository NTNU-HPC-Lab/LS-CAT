#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void move(uint8_t *buf, uint32_t dest, uint32_t source, uint16_t bytesEach, const bool wipe) {
extern __shared__ uint8_t sharedMemT[];
const uint32_t i = threadIdx.x;

uint8_t *src = &buf[source];
for (uint16_t j = 0; j < bytesEach; j++)
{
sharedMemT[(i*bytesEach) + j] = src[(i*bytesEach) + j];
if (wipe){
src[(i*bytesEach) + j] = 0;
}
}

__syncthreads();

uint8_t *d = &buf[dest];
for (uint16_t j = 0; j < bytesEach; j++)
{
d[(i*bytesEach) + j] = sharedMemT[(i*bytesEach) + j];
}
}