#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void bitonicSort2(int *inputArray, const unsigned int stage, const unsigned int passOfStage, const unsigned int width) {
int4 *theArray = (int4 *)inputArray;
size_t i = blockIdx.x * blockDim.x + threadIdx.x; // get_global_id(0);
int4 srcLeft, srcRight, mask;
int4 imask10 = make_int4(0, 0, -1, -1);
int4 imask11 = make_int4(0, -1, 0, -1);
const unsigned int dir = 0;
if (stage > 0) {
if (passOfStage > 0) // upper level pass, exchange between two fours
{
size_t r = 1 << (passOfStage - 1);
size_t lmask = r - 1;
size_t left = ((i >> (passOfStage - 1)) << passOfStage) + (i & lmask);
size_t right = left + r;

srcLeft = theArray[left];
srcRight = theArray[right];

// mask = srcLeft < srcRight;
mask.x = srcLeft.x < srcRight.x;
mask.y = srcLeft.y < srcRight.y;
mask.z = srcLeft.z < srcRight.z;
mask.w = srcLeft.w < srcRight.w;

// int4 imin = (srcLeft & mask) | (srcRight & ~mask);
int4 imin;
imin.x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
imin.y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
imin.z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
imin.w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);

// int4 imax = (srcLeft & ~mask) | (srcRight & mask);
int4 imax;
imax.x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
imax.y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
imax.z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
imax.w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);

if (((i >> (stage - 1)) & 1) ^ dir) {
theArray[left] = imin;
theArray[right] = imax;
} else {
theArray[right] = imin;
theArray[left] = imax;
}
} else // last pass, sort inside one four
{
srcLeft = theArray[i];
// srcRight = srcLeft.zwxy;
srcRight = make_int4(srcLeft.z, srcLeft.w, srcLeft.x, srcLeft.y);

// mask = (srcLeft < srcRight) ^ imask10;
mask.x = (srcLeft.x < srcRight.x) ^ imask10.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask10.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask10.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask10.w;

if (((i >> stage) & 1) ^ dir) {
// srcLeft = (srcLeft & mask) | (srcRight & ~mask);
srcLeft.x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
srcLeft.y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
srcLeft.z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
srcLeft.w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);

// srcRight = srcLeft.yxwz;
srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

// mask = (srcLeft < srcRight) ^ imask11;
mask.x = (srcLeft.x < srcRight.x) ^ imask11.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask11.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask11.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask11.w;

// theArray[i] = (srcLeft & mask) | (srcRight & ~mask);
theArray[i].x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
theArray[i].y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
theArray[i].z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
theArray[i].w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);
} else {
// srcLeft = (srcLeft & ~mask) | (srcRight & mask);
srcLeft.x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
srcLeft.y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
srcLeft.z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
srcLeft.w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);

// srcRight = srcLeft.yxwz;
srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

// mask = (srcLeft < srcRight) ^ imask11;
mask.x = (srcLeft.x < srcRight.x) ^ imask11.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask11.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask11.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask11.w;

// theArray[i] = (srcLeft & ~mask) | (srcRight & mask);
theArray[i].x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
theArray[i].y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
theArray[i].z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
theArray[i].w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);
}
}
} else // first stage, sort inside one four
{
int4 imask0 = make_int4(0, -1, -1, 0);
srcLeft = theArray[i];

// srcRight = srcLeft.yxwz;
srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

// mask = (srcLeft < srcRight) ^ imask0;
mask.x = (srcLeft.x < srcRight.x) ^ imask0.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask0.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask0.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask0.w;

if (dir) {
// srcLeft = (srcLeft & mask) | (srcRight & ~mask);
srcLeft.x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
srcLeft.y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
srcLeft.z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
srcLeft.w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);
} else {
// srcLeft = (srcLeft & ~mask) | (srcRight & mask);
srcLeft.x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
srcLeft.y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
srcLeft.z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
srcLeft.w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);
}

// srcRight = srcLeft.zwxy;
srcRight = make_int4(srcLeft.z, srcLeft.w, srcLeft.x, srcLeft.y);

// mask = (srcLeft < srcRight) ^ imask10;
mask.x = (srcLeft.x < srcRight.x) ^ imask10.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask10.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask10.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask10.w;

if ((i & 1) ^ dir) {
// srcLeft = (srcLeft & mask) | (srcRight & ~mask);
srcLeft.x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
srcLeft.y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
srcLeft.z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
srcLeft.w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);

// srcRight = srcLeft.yxwz;
srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

// mask = (srcLeft < srcRight) ^ imask11;
mask.x = (srcLeft.x < srcRight.x) ^ imask11.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask11.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask11.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask11.w;

// theArray[i] = (srcLeft & mask) | (srcRight & ~mask);
theArray[i].x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
theArray[i].y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
theArray[i].z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
theArray[i].w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);
} else {
// srcLeft = (srcLeft & ~mask) | (srcRight & mask);
srcLeft.x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
srcLeft.y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
srcLeft.z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
srcLeft.w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);

// srcRight = srcLeft.yxwz;
srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

// mask = (srcLeft < srcRight) ^ imask11;
mask.x = (srcLeft.x < srcRight.x) ^ imask11.x;
mask.y = (srcLeft.y < srcRight.y) ^ imask11.y;
mask.z = (srcLeft.z < srcRight.z) ^ imask11.z;
mask.w = (srcLeft.w < srcRight.w) ^ imask11.w;

// theArray[i] = (srcLeft & ~mask) | (srcRight & mask);
theArray[i].x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
theArray[i].y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
theArray[i].z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
theArray[i].w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);
}
}
}