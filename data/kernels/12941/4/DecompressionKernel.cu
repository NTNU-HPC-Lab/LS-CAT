#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DecompressionKernel(int dimensionalityd, unsigned char *compressed_data_buffer_in, int *chunk_boundaries_buffer_in, unsigned long long *uncompressed_data_buffer_out) {
register int offset, code, bcount, off, beg, end, lane, warp, iindex, lastidx, start, term;
register unsigned long long diff, prev;
__shared__ int ibufs[32 * (3 * WARPSIZE / 2)];


// index within this warp
lane = threadIdx.x & 31;
// index within shared prefix sum array
iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
ibufs[iindex] = 0;
iindex += WARPSIZE / 2;
lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
// warp id
warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
// prediction index within previous subchunk
offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

// determine start and end of chunk to decompress
start = 0;
if (warp > 0)
start = chunk_boundaries_buffer_in[warp - 1];
term = chunk_boundaries_buffer_in[warp];
off = ((start + 1) / 2 * 17);

prev = 0;
for (int i = start + lane; i < term; i += WARPSIZE) {
// read in half-bytes of size and leading-zero count information

if ((lane & 1) == 0) {
code = compressed_data_buffer_in[off + (lane >> 1)];

//4352
// printf(" %i ", start);
return;
ibufs[iindex] = code; //THIS line is crashing
return;
ibufs[iindex + 1] = code >> 4;

}
return;
off += (WARPSIZE / 2);
__threadfence_block();
code = ibufs[iindex];

bcount = code & 7;
if (bcount >= 2)
bcount++;

// calculate start positions of compressed data
ibufs[iindex] = bcount;
__threadfence_block();
ibufs[iindex] += ibufs[iindex - 1];
__threadfence_block();
ibufs[iindex] += ibufs[iindex - 2];
__threadfence_block();
ibufs[iindex] += ibufs[iindex - 4];
__threadfence_block();
ibufs[iindex] += ibufs[iindex - 8];
__threadfence_block();
ibufs[iindex] += ibufs[iindex - 16];
__threadfence_block();

// read in compressed data (the non-zero bytes)
beg = off + ibufs[iindex - 1];
off += ibufs[lastidx];
end = beg + bcount - 1;
diff = 0;
for (; beg <= end; end--) {
diff <<= 8;
diff |= compressed_data_buffer_in[end];
}

// negate delta if sign bit indicates it was negated during compression
if ((code & 8) != 0) {
diff = -diff;
}

// write out the uncompressed word
uncompressed_data_buffer_out[i] = prev + diff;
__threadfence_block();

// save prediction for next subchunk
prev = uncompressed_data_buffer_out[i + offset];
}
}