#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_add(int* big_set_numbers, const int big_set_count, int* tiny_set_numbers,const int tiny_set_count)		// __global__ prefix i vscc tarafindan anlasilmaz ve bu fonksiyonu nvcc compile edecektir
{
extern __shared__ int tiny_shared[];		// shared memory uzerinde depolanacak array.. toplam shared memory alani block sayisina bolunerek, her dilim bir block icin tahsis edilir..

int tidX = threadIdx.x;

if (tidX < tiny_set_count)					// threadid, tiny_set_count tan kucuk oldudgu surece extern yapili tiny_shared i doldur..
{
tiny_shared[tidX] = tiny_set_numbers[tidX];
}
// blockDim sayisi, tiny_set_count tan fazla olabilir ve fazlalik thread ler sonraki satirlara gecebilir..
__syncthreads();			// tum thread lerin bu satira gelmesi beklenir.. Yani __syncthreads() bir bariyer gorevi gorur
// tum thread ler __syncthreads() e gelmeden, hic bir thread sonraki satira gecemez..
// tum thread lerin shread memory uzerinde tiny_shared i doldurmasindan sonra shared memory alanina, tiny_shared icin erisim yap

int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < big_set_count)
{
int total = big_set_numbers[id];
for (int i = 0; i < tiny_set_count; i++)
{
total += tiny_shared[i];
}

big_set_numbers[id] *= total;
}
}