#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_add(int* big_set_numbers, const int big_set_count, int* tiny_set_numbers,const int tiny_set_count)				// __global__ prefix i vscc tarafindan anlasilmaz ve bu fonksiyonu nvcc compile edecektir
{
int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < big_set_count)
{
int total = big_set_numbers[id];
for (int i = 0; i < tiny_set_count; i++)
{
total += tiny_set_numbers[i];
}

big_set_numbers[id] *= total;
}
}