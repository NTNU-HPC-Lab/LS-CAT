#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void gpu_add(int* gpu_numbers, const int numberCount)				// __global__ prefix i vscc tarafindan anlasilmaz ve bu fonksiyonu nvcc compile edecektir
{
int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < numberCount)
gpu_numbers[id] *= 2;

}