#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate_updateGradInput_kernel( float *gradInput, float *error, float* targetCentroids, const float *centroids,const float *input, const float *outputIndice, const float* output, const float* gradOutput, int inputSize, int outputSize, int outputWindowSize, float c, float d, float e, float lr)
{
__shared__ float buffer[WINDOWGATE_THREADS+1];
unsigned int tx = threadIdx.x;
unsigned int k = blockIdx.x;
const float *gradOutput_k = gradOutput + outputWindowSize*k;
const float *output_k = output + outputWindowSize*k;
const float *input_k = input + inputSize*k;
float *gradInput_k = gradInput + inputSize*k;
float centroid = centroids[k];

// get gradient of centroid
buffer[tx] = 0;
for (unsigned int i=tx; i<outputWindowSize; i+=blockDim.x)
{
buffer[tx] += gradOutput_k[i]*output_k[i]*((float)(i+1) - centroid);
}

// add (reduce)
for (unsigned int stride = WINDOWGATE_THREADS >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}

if (tx == 0)
{
int outputIdx = outputIndice[k];
float gradCentroid = buffer[0]*c;
centroid -= (lr*gradCentroid);
centroid += outputIdx-1;
centroid /= (float)(outputSize);
targetCentroids[k] = centroid;
buffer[WINDOWGATE_THREADS] = centroid*(float)(inputSize);
}

__syncthreads();
float targetCentroid = buffer[WINDOWGATE_THREADS];

buffer[tx] = 0;
// target is a gaussian blur
for (int i=tx; i<inputSize; i+=blockDim.x)
{
float target = (float)(i+1)-targetCentroid;
target = d*expf(target*target*e);
float input = input_k[i];
// dot product of logProbInput and probTarget (NLL)
buffer[tx] -= logf(input + 0.0000001)*target;
// grad input w.r.t. NLL
gradInput_k[i] = -target/(input + 0.0000001);
}

// add (reduce)
for (unsigned int stride = WINDOWGATE_THREADS >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}

if (tx == 0)
error[k] = buffer[tx];
}