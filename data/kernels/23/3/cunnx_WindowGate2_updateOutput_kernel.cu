#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate2_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *inputIndice, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, int train)
{
__shared__ float buffer[WINDOWGATE2_THREADS+1];
unsigned int tx = threadIdx.x;
unsigned int k = blockIdx.x;
const float *input_k = input + inputSize*k;
float *output_k = output + outputWindowSize*k;

// get coordinate of centoid
buffer[tx] = 0;
for (unsigned int i=tx; i<inputSize; i+=blockDim.x)
buffer[tx] += input_k[i]*(float)(i+1);

// add (reduce)
for (unsigned int stride = WINDOWGATE2_THREADS >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}

if (tx == 0)
{
float centroid = buffer[0];

// make centroid a number between 0 and 1
centroid /= (float)(inputSize);

normalizedCentroids[k] = centroid;
if ( train )
{
centroid += noise[k];
centroid = fminf(fmaxf(0,centroid),1);
}
// align centroid to output
centroid *= (float)(outputSize);

float inputIdx = centroid/(float)(inputSize) - 0.5*(float)inputWindowSize;
float outputIdx = centroid - 0.5*(float)outputWindowSize;

// clip indices
inputIdx = fminf(inputIdx, inputSize-inputWindowSize+1);
inputIdx = fmaxf(inputIdx, 1);
outputIdx = fminf(outputIdx, outputSize-outputWindowSize+1);
outputIdx = fmaxf(outputIdx, 1);

inputIdx = ceilf(inputIdx);
outputIdx = ceilf(outputIdx);
// align centroid to outputWindow
centroid -= (outputIdx-1);

inputIndice[k] = (int)inputIdx;
outputIndice[k] = (int)outputIdx;
centroids[k] = centroid;

buffer[WINDOWGATE2_THREADS] = inputIdx;
}

__syncthreads();

float inputIdx = buffer[WINDOWGATE2_THREADS];
const float *inputWindow = input_k + (int)inputIdx;

for (int i=tx; i<outputWindowSize; i+=blockDim.x)
{
output_k[i] = inputWindow[(int)floorf(((float)i)/windowStride)];
}
}