#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, int numCases, int numOut) {
const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
const int tidx = ty * numCases + tx;

if (ty < numOut && tx < numCases) {
float v = 0;
for (int j = 0; j < numOut; j++) {
v += dE_dy_l[j * numCases + tx] * ((j == ty) - y_l[j * numCases + tx]);
}
v *= y_l[tidx];
dE_dx_l[tidx] = v;
}
}