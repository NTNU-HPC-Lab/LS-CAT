#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMultiplication__CudaKernel(int* in_tabA, int* in_tabB, int* out_tabC, int outTabWidth)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

//making sure that extra threads will do not any work
if (row < outTabWidth && col < outTabWidth)
{
int tmp_sum = 0;

//#pragma unroll
for (int i = 0; i < outTabWidth; i++)
{
tmp_sum += in_tabA[row * outTabWidth + i] * in_tabB[i * outTabWidth + col];
}
out_tabC[row * outTabWidth + col] = tmp_sum;
}
}