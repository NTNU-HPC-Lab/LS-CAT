#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
// iterators
int xx, yy;

// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
output = output + o*output_w*output_h;
input = input + i*input_w*input_h;

// For all output pixels...
for(yy = yy_start; yy < yy_end; yy+=yy_step) {
for(xx = xx_start; xx < xx_end; xx+=xx_step) {
// Compute the mean of the input image...
float *ptr_input = input + yy*dH*input_w + xx*dW;
float *ptr_output = output + yy*output_w + xx;
float sum = 0;
int kx, ky;
for(ky = 0; ky < kH; ky++) {
for(kx = 0; kx < kW; kx++)
sum += ptr_input[kx];
ptr_input += input_w; // next input line
}
// Update output
*ptr_output = sum/float(kW*kH);
}
}
}