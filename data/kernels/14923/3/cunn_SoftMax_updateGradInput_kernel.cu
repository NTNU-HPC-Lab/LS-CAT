#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim, int stride)
{
__shared__ float buffer[SOFTMAX_THREADS];
float *gradInput_k = gradInput + blockIdx.x*dim*stride + blockIdx.y;
float *output_k = output + blockIdx.x*dim*stride + blockIdx.y;
float *gradOutput_k = gradOutput + blockIdx.x*dim*stride + blockIdx.y;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

// sum?
buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
buffer[threadIdx.x] += gradOutput_k[i*stride] * output_k[i*stride];

__syncthreads();

// reduce
if (threadIdx.x == 0)
{
float sum_k = 0;
for (int i=0; i<blockDim.x; i++)
sum_k += buffer[i];
buffer[0] = sum_k;
}

__syncthreads();

float sum_k = buffer[0];
for (int i=i_start; i<i_end; i+=i_step)
gradInput_k[i*stride] = output_k[i*stride] * (gradOutput_k[i*stride] - sum_k);
}