#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int r = by * blockDim.y + ty;
int c = bx * blockDim.x + tx;
int dimC = numAColumns;

__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

float value = 0.0;

for (int i=0; i < (dimC-1)/TILE_WIDTH +1; i++){


if ((r < numCRows) && ((i*TILE_WIDTH + tx)< dimC)){
ds_A[ty][tx]=A[r*dimC + i*TILE_WIDTH + tx];
} else {
ds_A[ty][tx]=0.0;
}

if ((c < numCColumns) && ((i*TILE_WIDTH + ty)< dimC)){
ds_B[ty][tx]=B[(i*TILE_WIDTH + ty)*numBColumns + c];
} else {
ds_B[ty][tx]=0.0;
}

__syncthreads();

for (int j=0; j<TILE_WIDTH; j++){
value += ds_A[ty][j] * ds_B[j][tx];
}

__syncthreads();

}

if ((r < numCRows) && (c < numCColumns)){
C[r*numCColumns+c] = value;
}
}