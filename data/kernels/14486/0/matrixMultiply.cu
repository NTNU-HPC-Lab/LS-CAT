#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

// Compute C = A * B



__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {


//@@ Insert code to implement matrix multiplication here
int r = blockIdx.x * blockDim.x + threadIdx.x;
int c = blockIdx.y * blockDim.y + threadIdx.y;

if ((r < numCRows) && (c < numCColumns)){
float value = 0.0;

for (int i=0; i < numAColumns; i++){
value += A[r*numAColumns+i] * B[i*numBColumns+c];
}
C[r*numCColumns+c] = value;
}

}