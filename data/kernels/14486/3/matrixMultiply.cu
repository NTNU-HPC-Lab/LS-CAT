#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {

// which thread is this?
int tx = blockIdx.x*blockDim.x + threadIdx.x;
int ty = blockIdx.y*blockDim.y + threadIdx.y;

if ((tx < numCRows) && (ty < numCColumns)){
float value = 0.0f;
for (unsigned int i = 0; i < numAColumns; ++i){
value += A[tx*numAColumns + i] * B[i*numBColumns + ty];
}
C[tx*numCColumns + ty] = value;
}

}