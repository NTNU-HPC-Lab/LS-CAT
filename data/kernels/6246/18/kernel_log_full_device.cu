#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_log_full_device(int *x, int *out, unsigned int size, int epsilon) {
unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < size; i += stride) {
out[i] = (int) log((float) x[i] + epsilon);
}
}