#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_pow_grad_device(int *x, int power, int *grad, int *out, bool grad_is_scalar, unsigned int size) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < size; i += stride) {
out[i] = grad[(grad_is_scalar) ? 0 : i] * ((int) power) * ((int) powf((float) x[i], power - 1));
}
}