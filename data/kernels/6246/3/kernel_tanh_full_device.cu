#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_tanh_full_device(unsigned int size, int *x, int *out) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int stride = blockDim.x * gridDim.x;

/* tanh : R -> (-1,1)  which is 0 in the integers */
for (unsigned int i = idx; i < size; i += stride) {
out[i] = 0;
}
}