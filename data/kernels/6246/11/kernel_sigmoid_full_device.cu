#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sigmoid_full_device(unsigned int size, int *x, int *out) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < size; i += stride) {
out[i] = 1 / (1 + abs(x[i]));
}
}