#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
c[i] += a[i]*b[i];
}
}