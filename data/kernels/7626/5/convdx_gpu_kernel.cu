#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convdx_gpu_kernel(float *dx, float *dy, float *weights, const int S,const int outSize, const int inSize){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;
if(row < inSize && col < outSize){
//		printf("row %d, col %d, bias[col] %.2f\n", row, col,bias[col]);
for(int i = 0; i < S; ++i){
dx[row*outSize+col] +=dy[row* S + i ]*weights[col*S+i];
//		  printf("dy[%d] is %.1f,weight[%d] is %.1f\n", row*S+i,dy[row*S+i],col*S+i,weights[col*S+i]);
}
//		printf("conv dx %d is %3f\n",row*outSize+col, dx[row*outSize+col] );
}
}