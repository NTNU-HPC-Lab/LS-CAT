#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convdw_gpu_kernel(float *dw, float *dy, float *x, const int S,const int outSize, const int inSize){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;
if(row < inSize && col < outSize){
//		printf("row %d, col %d, bias[col] %.2f\n", row, col,bias[col]);
for(int i = 0; i < S; ++i){
dw[row*outSize+col] +=x[row+S*i ]*dy[i*outSize+col];
//		  printf("x[%d] is %.1f,dy[%d] is %.1f\n", row + S*i,x[row + S*i],i*S+row,dy[i*outSize+col]);
}
//  		printf("conv dw %d is %3f\n",row*outSize+col, dw[row*outSize+col] );
}
}