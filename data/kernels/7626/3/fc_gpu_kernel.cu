#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fc_gpu_kernel(float *y, float *x, float *weights, const int weightHeight,const int outSize, const int inSize){
//printf(x);
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;
//printf("row %d, col %d in fc.cu \n",row,col);
if(row < inSize && col < outSize){
//float acc = 0;
for(int i = 0; i < weightHeight; ++i){
y[row*outSize+col] +=x[row*weightHeight + i ]*weights[i*outSize+col];
//printf("x[%d] is %.1f,weight[%d] is %.1f\n", row*weightHeight+i,x[row*weightHeight+i],i*outSize+col,weights[i*outSize+col]);
}
//printf("acc is %3f, y %d is %3f\n",acc, row*outSize+col, y[row*outSize+col] );
}
}