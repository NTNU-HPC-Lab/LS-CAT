#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int z, unsigned int d)
{
return (NX*(NY*(NZ*(d-1)+z)+y)+x);
}
__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y, unsigned int z)
{
return NX*(NY*z + y)+x;
}
__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y, unsigned int z)
{
return NX*(NY*z + y)+x;
}
__global__ void gpu_init_equilibrium(double *f0, double *f1, double *h0, double *h1, double *temp0, double *temp1, double *r, double *c, double *u, double *v, double *w, double *ex, double *ey, double *ez, double*temp)
{
unsigned int y = blockIdx.y;
unsigned int z = blockIdx.z;
unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;

double rho    = r[gpu_scalar_index(x,y,z)];
double ux     = u[gpu_scalar_index(x,y,z)];
double uy     = v[gpu_scalar_index(x,y,z)];
double uz     = w[gpu_scalar_index(x,y,z)];
double charge = c[gpu_scalar_index(x,y,z)];
double Ex     = ex[gpu_scalar_index(x,y,z)];
double Ey     = ey[gpu_scalar_index(x,y,z)];
double Ez     = ez[gpu_scalar_index(x,y,z)];
double Temp   = temp[gpu_scalar_index(x,y,z)];

// load equilibrium
// feq_i  = w_i rho [1 + 3(ci . u) + (9/2) (ci . u)^2 - (3/2) (u.u)]
// feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u) + (1/2) (ci . 3u)^2]
// feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u){ 1 + (1/2) (ci . 3u) }]

// temporary variables
double w0r = w0*rho;
double wsr = ws*rho;
double war = wa*rho;
double wdr = wd*rho;

double w0c = w0*charge;
double wsc = ws*charge;
double wac = wa*charge;
double wdc = wd*charge;

double w0t = w0*Temp;
double wst = ws*Temp;
double wat = wa*Temp;
double wdt = wd*Temp;

double omusq   = 1.0 - 0.5*(ux*ux+uy*uy+uz*uz)/cs_square;
double omusq_c = 1.0 - 0.5*((ux + K*Ex)*(ux + K*Ex) + (uy + K*Ey)*(uy + K*Ey) + (uz + K*Ez)*(uz + K*Ez)) / cs_square;

double tux   = ux / cs_square / CFL;
double tuy   = uy / cs_square / CFL;
double tuz   = uz / cs_square / CFL;
double tux_c = (ux + K*Ex) / cs_square / CFL;
double tuy_c = (uy + K*Ey) / cs_square / CFL;
double tuz_c = (uz + K*Ez) / cs_square / CFL;

// zero weight
f0[gpu_field0_index(x,y,z)]      = w0r*(omusq);
h0[gpu_field0_index(x,y,z)]      = w0c*(omusq_c);
temp0[gpu_field0_index(x, y, z)] = w0t*(omusq);

// adjacent weight
// flow
double cidot3u = tux;
f1[gpu_fieldn_index(x,y,z,1)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = -tux;
f1[gpu_fieldn_index(x,y,z,2)]  = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy;
f1[gpu_fieldn_index(x,y,z,3)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = -tuy;
f1[gpu_fieldn_index(x,y,z,4)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = tuz;
f1[gpu_fieldn_index(x,y,z,5)] = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuz;
f1[gpu_fieldn_index(x,y,z,6)] = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

// charge
cidot3u = tux_c;
h1[gpu_fieldn_index(x,y,z,1)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux_c;
h1[gpu_fieldn_index(x,y,z,2)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c;
h1[gpu_fieldn_index(x,y,z,3)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy_c;
h1[gpu_fieldn_index(x,y,z,4)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz_c;
h1[gpu_fieldn_index(x,y,z,5)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuz_c;
h1[gpu_fieldn_index(x,y,z,6)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

// temperature
cidot3u = tux;
temp1[gpu_fieldn_index(x, y, z, 1)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux;
temp1[gpu_fieldn_index(x, y, z, 2)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy;
temp1[gpu_fieldn_index(x, y, z, 3)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy;
temp1[gpu_fieldn_index(x, y, z, 4)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz;
temp1[gpu_fieldn_index(x, y, z, 5)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuz;
temp1[gpu_fieldn_index(x, y, z, 6)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

// diagonal weight
// flow
cidot3u = tux+tuy;
f1[gpu_fieldn_index(x,y,z,7)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = -tuy-tux;
f1[gpu_fieldn_index(x,y,z,8)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = tux+tuz;
f1[gpu_fieldn_index(x,y,z,9)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = -tux-tuz;
f1[gpu_fieldn_index(x,y,z,10)] = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
cidot3u = tuz + tuy;
f1[gpu_fieldn_index(x,y,z,11)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy - tuz;
f1[gpu_fieldn_index(x,y,z,12)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuy;
f1[gpu_fieldn_index(x,y,z,13)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tux;
f1[gpu_fieldn_index(x,y,z,14)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuz;
f1[gpu_fieldn_index(x,y,z,15)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tux;
f1[gpu_fieldn_index(x,y,z,16)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tuz;
f1[gpu_fieldn_index(x,y,z,17)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tuy;
f1[gpu_fieldn_index(x,y,z,18)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

// charge
cidot3u = tux_c + tuy_c;
h1[gpu_fieldn_index(x, y, z, 7)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy_c - tux_c;
h1[gpu_fieldn_index(x, y, z, 8)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c + tuz_c;
h1[gpu_fieldn_index(x, y, z, 9)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 10)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c + tuz_c;
h1[gpu_fieldn_index(x, y, z, 11)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 12)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c - tuy_c;
h1[gpu_fieldn_index(x, y, z, 13)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c - tux_c;
h1[gpu_fieldn_index(x, y, z, 14)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 15)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz_c - tux_c;
h1[gpu_fieldn_index(x, y, z, 16)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 17)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz_c - tuy_c;
h1[gpu_fieldn_index(x, y, z, 18)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

// temperature
cidot3u = tux + tuy;
temp1[gpu_fieldn_index(x, y, z, 7)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy - tux;
temp1[gpu_fieldn_index(x, y, z, 8)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux + tuz;
temp1[gpu_fieldn_index(x, y, z, 9)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux - tuz;
temp1[gpu_fieldn_index(x, y, z, 10)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy + tuz;
temp1[gpu_fieldn_index(x, y, z, 11)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy - tuz;
temp1[gpu_fieldn_index(x, y, z, 12)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuy;
temp1[gpu_fieldn_index(x, y, z, 13)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tux;
temp1[gpu_fieldn_index(x, y, z, 14)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuz;
temp1[gpu_fieldn_index(x, y, z, 15)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tux;
temp1[gpu_fieldn_index(x, y, z, 16)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tuz;
temp1[gpu_fieldn_index(x, y, z, 17)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tuy;
temp1[gpu_fieldn_index(x, y, z, 18)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));


// 3d diagonal
//flow
cidot3u = tux + tuy + tuz;
f1[gpu_fieldn_index(x, y, z, 19)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tuy - tux - tuz;
f1[gpu_fieldn_index(x, y, z, 20)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux + tuy - tuz;
f1[gpu_fieldn_index(x, y, z, 21)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tux - tuy;
f1[gpu_fieldn_index(x, y, z, 22)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux + tuz - tuy;
f1[gpu_fieldn_index(x, y, z, 23)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tux - tuz;
f1[gpu_fieldn_index(x, y, z, 24)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy + tuz - tux;
f1[gpu_fieldn_index(x, y, z, 25)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuy - tuz;
f1[gpu_fieldn_index(x, y, z, 26)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

//charge
cidot3u = tux_c + tuy_c + tuz_c;
h1[gpu_fieldn_index(x, y, z, 19)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux_c -tuy_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 20)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c + tuy_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 21)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz_c - tux_c - tuy_c;
h1[gpu_fieldn_index(x, y, z, 22)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c + tuz_c - tuy_c;
h1[gpu_fieldn_index(x, y, z, 23)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c - tux_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 24)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy_c + tuz_c - tux_c;
h1[gpu_fieldn_index(x, y, z, 25)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux_c - tuy_c - tuz_c;
h1[gpu_fieldn_index(x, y, z, 26)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

//temperature
cidot3u = tux + tuy + tuz;
temp1[gpu_fieldn_index(x, y, z, 19)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = -tux - tuy - tuz;
temp1[gpu_fieldn_index(x, y, z, 20)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux + tuy - tuz;
temp1[gpu_fieldn_index(x, y, z, 21)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuz - tux - tuy;
temp1[gpu_fieldn_index(x, y, z, 22)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux + tuz - tuy;
temp1[gpu_fieldn_index(x, y, z, 23)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy - tux - tuz;
temp1[gpu_fieldn_index(x, y, z, 24)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tuy + tuz - tux;
temp1[gpu_fieldn_index(x, y, z, 25)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
cidot3u = tux - tuy - tuz;
temp1[gpu_fieldn_index(x, y, z, 26)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
}