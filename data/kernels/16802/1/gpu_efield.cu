#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y, unsigned int z)
{
return NX*(NY*z + y)+x;
}
__global__ void gpu_efield(double *fi, double *ex, double *ey, double *ez){

unsigned int y = blockIdx.y;
unsigned int z = blockIdx.z;
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int xp1 = (x + 1) % NX;
unsigned int yp1 = (y + 1) % NY;
unsigned int zp1 = (z + 1) % NZ;

unsigned int xm1 = (NX + x - 1) % NX;
unsigned int ym1 = (NY + y - 1) % NY;
unsigned int zm1 = (NZ + z - 1) % NZ;

ex[gpu_scalar_index(x, y, z)] = 0.5*(fi[gpu_scalar_index(xm1,y,z)] - fi[gpu_scalar_index(xp1, y, z)]) / dx;
ey[gpu_scalar_index(x, y, z)] = 0.5*(fi[gpu_scalar_index(x, ym1, z)] - fi[gpu_scalar_index(x, yp1, z)]) / dy;
ez[gpu_scalar_index(x, y, z)] = 0.5*(fi[gpu_scalar_index(x, y, zm1)] - fi[gpu_scalar_index(x, y, zp1)]) / dz;
}