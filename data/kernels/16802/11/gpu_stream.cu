#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int z, unsigned int d)
{
return (NX*(NY*(NZ*(d-1)+z)+y)+x);
}
__global__ void gpu_stream(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *temp0, double *temp1, double *temp2)
{
unsigned int y = blockIdx.y;
unsigned int z = blockIdx.z;
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

// streaming step
unsigned int xp1 = (x + 1) % NX;
unsigned int yp1 = (y + 1) % NY;
unsigned int zp1 = (z + 1) % NZ;
unsigned int xm1 = (NX + x - 1) % NX;
unsigned int ym1 = (NY + y - 1) % NY;
unsigned int zm1 = (NZ + z - 1) % NZ;
// direction numbering scheme
// 6 2 5
// 3 0 1
// 7 4 8

// load populations from adjacent nodes (ft is post-streaming population of f1)
// flows
f1[gpu_fieldn_index(x, y, z, 1)] = f2[gpu_fieldn_index(xm1, y, z, 1)];
f1[gpu_fieldn_index(x, y, z, 2)] = f2[gpu_fieldn_index(xp1, y, z, 2)];
f1[gpu_fieldn_index(x, y, z, 3)] = f2[gpu_fieldn_index(x, ym1, z, 3)];
f1[gpu_fieldn_index(x, y, z, 4)] = f2[gpu_fieldn_index(x, yp1, z, 4)];
f1[gpu_fieldn_index(x, y, z, 5)] = f2[gpu_fieldn_index(x, y, zm1, 5)];
f1[gpu_fieldn_index(x, y, z, 6)] = f2[gpu_fieldn_index(x, y, zp1, 6)];
f1[gpu_fieldn_index(x, y, z, 7)] = f2[gpu_fieldn_index(xm1, ym1, z, 7)];
f1[gpu_fieldn_index(x, y, z, 8)] = f2[gpu_fieldn_index(xp1, yp1, z, 8)];
f1[gpu_fieldn_index(x, y, z, 9)] = f2[gpu_fieldn_index(xm1, y, zm1, 9)];
f1[gpu_fieldn_index(x, y, z, 10)] = f2[gpu_fieldn_index(xp1, y, zp1, 10)];
f1[gpu_fieldn_index(x, y, z, 11)] = f2[gpu_fieldn_index(x, ym1, zm1, 11)];
f1[gpu_fieldn_index(x, y, z, 12)] = f2[gpu_fieldn_index(x, yp1, zp1, 12)];
f1[gpu_fieldn_index(x, y, z, 13)] = f2[gpu_fieldn_index(xm1, yp1, z, 13)];
f1[gpu_fieldn_index(x, y, z, 14)] = f2[gpu_fieldn_index(xp1, ym1, z, 14)];
f1[gpu_fieldn_index(x, y, z, 15)] = f2[gpu_fieldn_index(xm1, y, zp1, 15)];
f1[gpu_fieldn_index(x, y, z, 16)] = f2[gpu_fieldn_index(xp1, y, zm1, 16)];
f1[gpu_fieldn_index(x, y, z, 17)] = f2[gpu_fieldn_index(x, ym1, zp1, 17)];
f1[gpu_fieldn_index(x, y, z, 18)] = f2[gpu_fieldn_index(x, yp1, zm1, 18)];
f1[gpu_fieldn_index(x, y, z, 19)] = f2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
f1[gpu_fieldn_index(x, y, z, 20)] = f2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
f1[gpu_fieldn_index(x, y, z, 21)] = f2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
f1[gpu_fieldn_index(x, y, z, 22)] = f2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
f1[gpu_fieldn_index(x, y, z, 23)] = f2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
f1[gpu_fieldn_index(x, y, z, 24)] = f2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
f1[gpu_fieldn_index(x, y, z, 25)] = f2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
f1[gpu_fieldn_index(x, y, z, 26)] = f2[gpu_fieldn_index(xm1, yp1, zp1, 26)];

// charges
h1[gpu_fieldn_index(x, y, z, 1)] = h2[gpu_fieldn_index(xm1, y, z, 1)];
h1[gpu_fieldn_index(x, y, z, 2)] = h2[gpu_fieldn_index(xp1, y, z, 2)];
h1[gpu_fieldn_index(x, y, z, 3)] = h2[gpu_fieldn_index(x, ym1, z, 3)];
h1[gpu_fieldn_index(x, y, z, 4)] = h2[gpu_fieldn_index(x, yp1, z, 4)];
h1[gpu_fieldn_index(x, y, z, 5)] = h2[gpu_fieldn_index(x, y, zm1, 5)];
h1[gpu_fieldn_index(x, y, z, 6)] = h2[gpu_fieldn_index(x, y, zp1, 6)];
h1[gpu_fieldn_index(x, y, z, 7)] = h2[gpu_fieldn_index(xm1, ym1, z, 7)];
h1[gpu_fieldn_index(x, y, z, 8)] = h2[gpu_fieldn_index(xp1, yp1, z, 8)];
h1[gpu_fieldn_index(x, y, z, 9)] = h2[gpu_fieldn_index(xm1, y, zm1, 9)];
h1[gpu_fieldn_index(x, y, z, 10)] = h2[gpu_fieldn_index(xp1, y, zp1, 10)];
h1[gpu_fieldn_index(x, y, z, 11)] = h2[gpu_fieldn_index(x, ym1, zm1, 11)];
h1[gpu_fieldn_index(x, y, z, 12)] = h2[gpu_fieldn_index(x, yp1, zp1, 12)];
h1[gpu_fieldn_index(x, y, z, 13)] = h2[gpu_fieldn_index(xm1, yp1, z, 13)];
h1[gpu_fieldn_index(x, y, z, 14)] = h2[gpu_fieldn_index(xp1, ym1, z, 14)];
h1[gpu_fieldn_index(x, y, z, 15)] = h2[gpu_fieldn_index(xm1, y, zp1, 15)];
h1[gpu_fieldn_index(x, y, z, 16)] = h2[gpu_fieldn_index(xp1, y, zm1, 16)];
h1[gpu_fieldn_index(x, y, z, 17)] = h2[gpu_fieldn_index(x, ym1, zp1, 17)];
h1[gpu_fieldn_index(x, y, z, 18)] = h2[gpu_fieldn_index(x, yp1, zm1, 18)];
h1[gpu_fieldn_index(x, y, z, 19)] = h2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
h1[gpu_fieldn_index(x, y, z, 20)] = h2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
h1[gpu_fieldn_index(x, y, z, 21)] = h2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
h1[gpu_fieldn_index(x, y, z, 22)] = h2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
h1[gpu_fieldn_index(x, y, z, 23)] = h2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
h1[gpu_fieldn_index(x, y, z, 24)] = h2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
h1[gpu_fieldn_index(x, y, z, 25)] = h2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
h1[gpu_fieldn_index(x, y, z, 26)] = h2[gpu_fieldn_index(xm1, yp1, zp1, 26)];

// temperature
temp1[gpu_fieldn_index(x, y, z, 1)] = temp2[gpu_fieldn_index(xm1, y, z, 1)];
temp1[gpu_fieldn_index(x, y, z, 2)] = temp2[gpu_fieldn_index(xp1, y, z, 2)];
temp1[gpu_fieldn_index(x, y, z, 3)] = temp2[gpu_fieldn_index(x, ym1, z, 3)];
temp1[gpu_fieldn_index(x, y, z, 4)] = temp2[gpu_fieldn_index(x, yp1, z, 4)];
temp1[gpu_fieldn_index(x, y, z, 5)] = temp2[gpu_fieldn_index(x, y, zm1, 5)];
temp1[gpu_fieldn_index(x, y, z, 6)] = temp2[gpu_fieldn_index(x, y, zp1, 6)];
temp1[gpu_fieldn_index(x, y, z, 7)] = temp2[gpu_fieldn_index(xm1, ym1, z, 7)];
temp1[gpu_fieldn_index(x, y, z, 8)] = temp2[gpu_fieldn_index(xp1, yp1, z, 8)];
temp1[gpu_fieldn_index(x, y, z, 9)] = temp2[gpu_fieldn_index(xm1, y, zm1, 9)];
temp1[gpu_fieldn_index(x, y, z, 10)] = temp2[gpu_fieldn_index(xp1, y, zp1, 10)];
temp1[gpu_fieldn_index(x, y, z, 11)] = temp2[gpu_fieldn_index(x, ym1, zm1, 11)];
temp1[gpu_fieldn_index(x, y, z, 12)] = temp2[gpu_fieldn_index(x, yp1, zp1, 12)];
temp1[gpu_fieldn_index(x, y, z, 13)] = temp2[gpu_fieldn_index(xm1, yp1, z, 13)];
temp1[gpu_fieldn_index(x, y, z, 14)] = temp2[gpu_fieldn_index(xp1, ym1, z, 14)];
temp1[gpu_fieldn_index(x, y, z, 15)] = temp2[gpu_fieldn_index(xm1, y, zp1, 15)];
temp1[gpu_fieldn_index(x, y, z, 16)] = temp2[gpu_fieldn_index(xp1, y, zm1, 16)];
temp1[gpu_fieldn_index(x, y, z, 17)] = temp2[gpu_fieldn_index(x, ym1, zp1, 17)];
temp1[gpu_fieldn_index(x, y, z, 18)] = temp2[gpu_fieldn_index(x, yp1, zm1, 18)];
temp1[gpu_fieldn_index(x, y, z, 19)] = temp2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
temp1[gpu_fieldn_index(x, y, z, 20)] = temp2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
temp1[gpu_fieldn_index(x, y, z, 21)] = temp2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
temp1[gpu_fieldn_index(x, y, z, 22)] = temp2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
temp1[gpu_fieldn_index(x, y, z, 23)] = temp2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
temp1[gpu_fieldn_index(x, y, z, 24)] = temp2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
temp1[gpu_fieldn_index(x, y, z, 25)] = temp2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
temp1[gpu_fieldn_index(x, y, z, 26)] = temp2[gpu_fieldn_index(xm1, yp1, zp1, 26)];
}