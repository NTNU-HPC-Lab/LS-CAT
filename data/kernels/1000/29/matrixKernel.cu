#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixKernel(float* d_in, float* d_out) {
// Block index
int bx = blockIdx.x;
int by = blockIdx.y;

// Thread index (current coefficient)
int tx = threadIdx.x;
int ty = threadIdx.y;

float dividend =
d_in[(by * BLOCK_SIZE + 0) * STRIDE + (bx * BLOCK_SIZE + 0)];
float divisor =
d_in[(by * BLOCK_SIZE + ty) * STRIDE + (bx * BLOCK_SIZE + tx)];

d_out[(by * BLOCK_SIZE + ty) * STRIDE + (bx * BLOCK_SIZE + tx)] =
dividend / divisor;
}