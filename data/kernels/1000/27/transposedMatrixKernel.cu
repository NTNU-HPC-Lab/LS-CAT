#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposedMatrixKernel(int* d_a, int* d_b) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;

d_b[i * N + j] = d_a[j * N + i];
}