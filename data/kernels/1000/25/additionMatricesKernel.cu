#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

while (i < N) {
j = threadIdx.y + blockIdx.y * blockDim.y;
while (j < N) {
d_c[i * N + j] = d_a[i * N + j] + d_b[i * N + j];
j += blockDim.y * gridDim.y;
}
i += blockDim.x * gridDim.x;
}
}