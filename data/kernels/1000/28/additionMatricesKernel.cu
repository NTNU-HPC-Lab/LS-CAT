#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

d_c[i * N + j] = d_a[i * N + j] + d_b[i * N + j];
}