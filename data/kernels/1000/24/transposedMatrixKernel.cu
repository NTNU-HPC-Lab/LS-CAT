#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposedMatrixKernel(int* d_a, int* d_b) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;

while (i < N) {
j = threadIdx.y + blockDim.y * blockIdx.y;
while (j < N) {
d_b[i * N + j] = d_a[j * N + i];
j += blockDim.y * gridDim.y;
}
i += blockDim.x * gridDim.x;
}
}