#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MotionVec(float *new_image_dev, float *old_image_dev, uchar4 *Image_dev, int w, int h )
{
const int ix = blockDim.x * blockIdx.x + threadIdx.x;
const int iy = blockDim.y * blockIdx.y + threadIdx.y;
const float x = (float)ix + 0.5f;
const float y = (float)iy + 0.5f;
float diff = 0;

diff = old_image_dev[w*iy + ix] - new_image_dev[w*iy + ix];
diff *= diff;

float threshold = 5000;

if (diff > threshold)
{
Image_dev[w*iy + ix].x = 0;			//B  /* MODIFY CODE HERE*/
Image_dev[w*iy + ix].y = 0;			//G  /* MODIFY CODE HERE*/
Image_dev[w*iy + ix].z = 255;		//R  /* MODIFY CODE HERE*/
}
}