#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuParallelSqrt(float *dist, int width, int k){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
dist[yIndex*width + xIndex] = sqrt(dist[yIndex*width + xIndex]);
}