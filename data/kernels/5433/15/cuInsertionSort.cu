#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k){

// Variables
int l, i, j;
float *p_dist;
long  *p_ind;
float curr_dist, max_dist;
long  curr_row,  max_row;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){
// Pointer shift, initialization, and max value
p_dist   = dist + xIndex;
p_ind    = ind  + xIndex;
max_dist = p_dist[0];
p_ind[0] = 1;

// Part 1 : sort kth firt elementZ
for (l=1; l<k; l++){
curr_row  = l * width;
curr_dist = p_dist[curr_row];
if (curr_dist<max_dist){
i=l-1;
for (int a=0; a<l-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=l; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
} else {
p_ind[l*width] = l+1;
}
max_dist = p_dist[curr_row];
}

// Part 2 : insert element in the k-th first lines
max_row = (k-1)*width;
for (l=k; l<height; l++){
curr_dist = p_dist[l*width];
if (curr_dist<max_dist){
i=k-1;
for (int a=0; a<k-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=k-1; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
max_dist             = p_dist[max_row];
}
}
}
}