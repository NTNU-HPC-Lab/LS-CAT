#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void rowNorm_a( float* X, float* v, float* a, unsigned int size, unsigned int n)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;
unsigned int row;

for (unsigned long int i = idx; i < size; i += stride) {
row = (int)i/n;
X[i] /= v[row]*a[row];
}
}