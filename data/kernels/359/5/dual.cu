#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void dual( float* err, float* Y, float* X, float* Z, unsigned int size)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;
float temp;

err[idx] = 0.0;

for (unsigned int i = idx; i < size; i += stride) {
temp = X[i] - Z[i];
Y[i] += temp;
err[idx] += temp*temp;
}
//    __syncthreads();
}