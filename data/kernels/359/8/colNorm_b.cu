#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void colNorm_b( float* X, float* v, float* b, unsigned int size, unsigned int n)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;
unsigned int col;

for (unsigned long int i = idx; i < size; i += stride) {
col = (int)i%n;
X[i] /= v[col]*b[col];
}
}