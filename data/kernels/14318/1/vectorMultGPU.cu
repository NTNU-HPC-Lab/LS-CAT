#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorMultGPU(float *a, float *b, float *c, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

while (i < n)
{
c[i] = a[i] * b[i];
i+= blockDim.x * gridDim.x;
}
}