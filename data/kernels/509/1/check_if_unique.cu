#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_if_unique(const unsigned *keys, unsigned       *is_unique, size_t          kSize) {
unsigned id = threadIdx.x +
blockIdx.x * blockDim.x +
blockIdx.y * blockDim.x * gridDim.x;
if (id == 0) {
is_unique[0] = 1;
} else if (id < kSize) {
is_unique[id] = (keys[id] != keys[id - 1] ? 1 : 0);
}
}