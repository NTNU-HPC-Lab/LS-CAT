#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = i + offset;

if (k + blockDim.x < n)
{
C[k]            = A[i]            + B[i];
C[k + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
}
}