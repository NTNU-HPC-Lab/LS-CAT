#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
unsigned int k = i + offset;

if (k < n) C[i] = A[k]                  + B[k];
if (k + blockDim.x < n) {
C[i + blockDim.x]     = A[k + blockDim.x]     + B[k + blockDim.x];
}
if (k + 2 * blockDim.x < n) {
C[i + 2 * blockDim.x] = A[k + 2 * blockDim.x] + B[k + 2 * blockDim.x];
}
if (k + 3 * blockDim.x < n) {
C[i + 3 * blockDim.x] = A[k + 3 * blockDim.x] + B[k + 3 * blockDim.x];
}
}