#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_add_wavelet ( float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
// global grid idx for (x,y) plane
int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);
unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int idx = ipos * nx + ix;

if(ix == nx / 2) g_u2[idx] += wavelets;
}