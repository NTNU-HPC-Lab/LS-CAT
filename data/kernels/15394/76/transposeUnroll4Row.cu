#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

unsigned int ti = iy * nx + ix; // access in rows
unsigned int to = ix * ny + iy; // access in columns

if (ix + 3 * blockDim.x < nx && iy < ny)
{
out[to]                   = in[ti];
out[to + ny * blockDim.x]   = in[ti + blockDim.x];
out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
}
}