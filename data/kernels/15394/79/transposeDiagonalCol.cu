#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
unsigned int blk_y = blockIdx.x;
unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

unsigned int ix = blockDim.x * blk_x + threadIdx.x;
unsigned int iy = blockDim.y * blk_y + threadIdx.y;

if (ix < nx && iy < ny)
{
out[iy * nx + ix] = in[ix * ny + iy];
}
}