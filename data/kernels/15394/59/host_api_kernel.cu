#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int nthreads = gridDim.x * blockDim.x;

for (i = tid; i < N; i += nthreads)
{
float rand = randomValues[i];
rand = rand * 2;
out[i] = rand;
}
}