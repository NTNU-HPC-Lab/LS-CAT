#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Find BLANK and replace your own code.
* And submit report why do you replace the blank that way.
*/

/* 2015004693_YangSangheon */


#define TILE_WIDTH 24      /* set TILE_WIDTH 16 for the evaluation! */
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;




__global__ void gemm(float *a, float *b, float *c, const float alpha, const float beta, float *output, const int input_size){
// a, b, c : input matrix address
// alpha, beta : input constant
// output : output buffer address
// input_size : width, height of input matrix
// all input, output matrices are vectorized

int tx = threadIdx.x, ty = threadIdx.y;
int bx = blockIdx.x,  by = blockIdx.y;

int row = by*blockDim.y + ty;
int col = bx*blockDim.x + tx;

//if(row>=input_size ||col>=input_size) { return; }

if(row >= (input_size/TILE_WIDTH+1)*TILE_WIDTH ||col >= (input_size/TILE_WIDTH+1)*TILE_WIDTH) {return;}

// allocate 2D tiles in __shared__ memory
__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

float result = 0;

// make sure you handle the case when the matrix sizes are not
// multiple of TILE_WIDTH!
// loop over the tiles of the input in phases

int a_index;
int b_index;

for(int p = 0; p < input_size/TILE_WIDTH+1 ;p++){
// CHANGE
// You need to use __syncthreads() a few times
// to synchronize the threads in a thread block.
a_index = row*input_size + p*TILE_WIDTH +tx;
b_index = (ty + p*TILE_WIDTH)*input_size + col;

if(a_index < input_size * input_size )
s_a[ty][tx] = a[a_index];
else
s_a[ty][tx] = 0.0;

if(b_index < input_size*input_size )
s_b[ty][tx] = b[b_index];
else
s_b[ty][tx] = 0.0;

//		s_a[ty][tx] = a[row*input_size + p*TILE_WIDTH+tx];
//		s_b[ty][tx] = b[(ty+p*TILE_WIDTH)*input_size + col];

__syncthreads();

for(int i = 0; i<TILE_WIDTH; i++)
result += s_a[ty][i] * s_b[i][tx];

__syncthreads();

}
//__syncthreads();
// write out the result to output[row*input_size + col]
// CHANGE
if(row < input_size && col < input_size)
output[row*input_size + col] = (alpha * result) + (beta * c[row*input_size + col]);
//__syncthreads();
}