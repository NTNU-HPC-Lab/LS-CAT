#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Find BLANK and replace your own code.
* And submit report why do you replace the blank that way.
*/

/* 2015004693_YangSangheon */


#define TILE_WIDTH 24      /* set TILE_WIDTH 16 for the evaluation! */
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;




__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
// input : input_matrix address
// output : output buffer address
// input_size : width, height of input matrix
// filter_size : filter_size of maxpolling
// all input, output matrices are vectorized

int col = blockDim.x * blockIdx.x + threadIdx.x;
int row = blockDim.y * blockIdx.y + threadIdx.y;

// out of bound
// CHANGE

float tmp = 0.0;
float Max = -999999.9;

for(int i = 0; i < filter_size; i++){
for(int j = 0; j < filter_size; j++){
tmp = input[(input_size*filter_size*row)+(filter_size*col)+(input_size*j)+i];
if(Max<tmp)
Max = tmp;
}
}

if(col < (input_size/filter_size) && row < (input_size/filter_size))
output[((input_size/filter_size)*row)+col] = Max;

//printf("thread_made\n");
}