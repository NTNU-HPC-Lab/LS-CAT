#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void optimalTransposeKernel(const float *input, float *output, int n)
{
__shared__ float tile[64][65];
int x = blockIdx.x * 64 + threadIdx.x;
int y = blockIdx.y * 64 + threadIdx.y;
const int width = gridDim.x * 64;
const int height = gridDim.y * 64;
if (x < width && y < height)
{ tile[threadIdx.y][threadIdx.x] = input[y*width + x];
tile[threadIdx.y+16][threadIdx.x] = input[(y+16)*width +x];
tile[threadIdx.y+32][threadIdx.x] = input[(y+32)*width +x];
tile[threadIdx.y+48][threadIdx.x] = input[(y+48)*width +x];
}
__syncthreads();

x = blockIdx.y * 64 + threadIdx.x; // transpose block offset
y = blockIdx.x * 64 + threadIdx.y;
if (y < width && x < height)
{ output[y*height + x] = tile[threadIdx.x][threadIdx.y];
output[(y+16)*height +x] = tile[threadIdx.x][threadIdx.y+16];
output[(y+32)*height +x] = tile[threadIdx.x][threadIdx.y+32];
output[(y+48)*height +x] = tile[threadIdx.x][threadIdx.y+48];
}
}