#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Read TODO items below
*/




__global__
__global__ void cacheMatmul(float *a, float *b, float *c, int n)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

float acc = 0;
for(int k1=0;k1<n;k1+=gridDim.x)
{
acc=c[i*n+j];
for(int k=k1;k<k1+gridDim.x;k++)
{
acc += a[i*n+k] * b[k*n+j];
}
c[i*n+j] = acc;
}
}