#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Read TODO items below
*/




__global__
__global__ void sharedMatmul(float *a, float *b, float *c, int n)
{

__shared__ float A_tile[32][32];
__shared__ float B_tile[32][32];
int width = gridDim.x*blockDim.x;

float acc = 0;

int i = blockIdx.x*32 + threadIdx.x;
int j = blockIdx.y*32 + threadIdx.y;


/* Accumulate C tile by tile. */

for (int tileIdx = 0; tileIdx < gridDim.x ; tileIdx+=1)
{

/* Load one tile of A and one tile of B into shared mem */

A_tile[threadIdx.y][ threadIdx.x] = a[j * width + tileIdx*32+threadIdx.x];
B_tile[threadIdx.y][threadIdx.x] = b[(tileIdx * 32 + threadIdx.y)* width+ i ];

__syncthreads();

/* Accumulate one tile of C from tiles of A and B in shared mem */

for (int k = 0 ;k < 32; k++)
{
acc += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
}

__syncthreads();

}

c[j * width + i ] = acc;

}