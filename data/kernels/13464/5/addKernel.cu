#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
for(long i=0;i<1024*500;i++){
c[i] = a[i]*10 + b[i] * 5;
}
//printf("addKernel::threadIdx: %d, %d, %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}