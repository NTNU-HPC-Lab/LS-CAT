#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReducePI2( float* d_sum, int num, float* d_pi ){
int id=threadIdx.x;
extern float __shared__ s_sum[];
s_sum[id]=d_sum[id];
__syncthreads();
for(int i=(blockDim.x>>1);i>0;i>>=1){
if(id<i)
s_sum[id]+=s_sum[id+i];
__syncthreads();
}
printf("%d,%f\n",id,s_sum[id]);
if(id==0){
*d_pi=s_sum[0]/num;
printf("%d,%f\n",id,*d_pi);
}

}