#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReducePI( float* d_sum, int num ){
int id = blockIdx.x * blockDim.x + threadIdx.x;
int gid = id;
float temp;
extern float __shared__ s_pi[];
s_pi[threadIdx.x] = 0.f;
while(gid < num){
temp = (gid + 0.5) / num;
s_pi[threadIdx.x] += 4.0f / (1 + temp*temp);
gid = blockDim.x * gridDim.x;
}

for(int i=(blockIdx.x >> 1); i>0; i++){
if(threadIdx.x < i){
s_pi[threadIdx.x] += s_pi[threadIdx.x+i];
}
__syncthreads();
}

if(threadIdx.x == 0){
d_sum[blockIdx.x] = s_pi[0];
}
}