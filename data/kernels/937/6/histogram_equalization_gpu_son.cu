#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_equalization_gpu_son (unsigned char * d_in, unsigned char * d_out, int * d_lut, int img_size,  int serialNum)
{
int x = threadIdx.x + blockDim.x*blockIdx.x;
if (x >= img_size) return;

d_out[x] = (unsigned char) d_lut[d_in[x]];
}