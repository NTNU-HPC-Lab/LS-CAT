#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void rgb2yuv_gpu_son(unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, unsigned char * d_y , unsigned char * d_u ,unsigned char * d_v , int size)
{
int x = threadIdx.x + blockDim.x*blockIdx.x;
if (x >= size) return;
unsigned char r, g, b;

r = d_r[x];
g = d_g[x];
b = d_b[x];

d_y[x] = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
d_u[x] = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
d_v[x] = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
}