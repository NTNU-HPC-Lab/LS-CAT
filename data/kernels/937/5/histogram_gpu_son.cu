#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_gpu_son(unsigned char * d_img, unsigned int * d_hist,  int img_size,  int serialNum)
{
// __shared__ unsigned int aa[ROLLSIZE][256];
extern __shared__ unsigned int aa[];
int x = threadIdx.x + blockDim.x*blockIdx.x;
int i;

for(i = 0; i < ROLLSIZE; i++) aa[(i << 8) + threadIdx.x] = 0;
__syncthreads();

int end = (x+1)*serialNum;
if (end >= img_size) end = img_size;

for(i = x*serialNum; i < end; i++) atomicAdd(&(aa[((threadIdx.x >> 4 ) << 8) +  d_img[i]]), 1);
__syncthreads();

unsigned int s;
for(s = 16 / 2; s > 0; s >>= 1) {
//Only when numThreads == 256
for(i = 0; i < s; i++) aa[(i << 8) + threadIdx.x] += aa[((i+s) << 8) + threadIdx.x];


// if (threadIdx.x < s) {
// for(i = 0; i < 256; i++) {
//     aa[threadIdx.x][i] += aa[threadIdx.x + s][i];
// }
// }
__syncthreads();
}

atomicAdd(&(d_hist[threadIdx.x]),aa[threadIdx.x]);
return;
}