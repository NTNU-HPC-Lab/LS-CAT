#include "hip/hip_runtime.h"
#include "includes.h"


__device__ float Hue_2_RGB_gpu( float v1, float v2, float vH )             //Function Hue_2_RGB
{
if ( vH < 0 ) vH += 1;
if ( vH > 1 ) vH -= 1;
if ( ( 6 * vH ) < 1 ) return ( v1 + ( v2 - v1 ) * 6 * vH );
if ( ( 2 * vH ) < 1 ) return ( v2 );
if ( ( 3 * vH ) < 2 ) return ( v1 + ( v2 - v1 ) * ( ( 2.0f/3.0f ) - vH ) * 6 );
return ( v1 );
}
__global__ void hsl2rgb_gpu_son(float * d_h , float * d_s ,unsigned char * d_l , unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, int size)
{
int x = threadIdx.x + blockDim.x*blockIdx.x;
if (x >= size) return;
float H = d_h[x];
float S = d_s[x];
float L = d_l[x]/255.0f;
float var_1, var_2;
unsigned char r,g,b;

if ( S == 0 )
{
r = L * 255;
g = L * 255;
b = L * 255;
}
else
{

if ( L < 0.5 )
var_2 = L * ( 1 + S );
else
var_2 = ( L + S ) - ( S * L );

var_1 = 2 * L - var_2;
r = 255 * Hue_2_RGB_gpu( var_1, var_2, H + (1.0f/3.0f) );
g = 255 * Hue_2_RGB_gpu( var_1, var_2, H );
b = 255 * Hue_2_RGB_gpu( var_1, var_2, H - (1.0f/3.0f) );
}
d_r[x] = r;
d_g[x] = g;
d_b[x] = b;
}