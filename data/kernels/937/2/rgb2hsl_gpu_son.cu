#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void rgb2hsl_gpu_son( unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, float * d_h , float * d_s , unsigned char * d_l , int size)
{
int x = threadIdx.x + blockDim.x*blockIdx.x;
if (x >= size) return;

float H,S,L;
float var_r = ( (float)d_r[x]/255 );//Convert RGB to [0,1]
float var_g = ( (float)d_g[x]/255 );
float var_b = ( (float)d_b[x]/255 );
float var_min = (var_r < var_g) ? var_r : var_g;
var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
float var_max = (var_r > var_g) ? var_r : var_g;
var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
float del_max = var_max - var_min;               //Delta RGB value

L = ( var_max + var_min ) / 2;
if ( del_max == 0 )//This is a gray, no chroma...
{
H = 0;
S = 0;
}
else                                    //Chromatic data...
{
if ( L < 0.5 )
S = del_max/(var_max+var_min);
else {
S = del_max/(2-var_max-var_min );

float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
if( var_r == var_max ) H = del_b - del_g;
else {
if( var_g == var_max ) H = (1.0/3.0) + del_r - del_b;
else H = (2.0/3.0) + del_g - del_r;
}
}

if ( H < 0 ) H += 1;
if ( H > 1 ) H -= 1;
}
d_h[x] = H;
d_s[x] = S;
d_l[x] = (unsigned char)(L*255);
}