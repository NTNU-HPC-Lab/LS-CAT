#include "hip/hip_runtime.h"
#include "includes.h"


__device__ unsigned char clip_rgb_gpu(int x)
{
if(x > 255)
return 255;
if(x < 0)
return 0;

return (unsigned char)x;
}
__global__ void yuv2rgb_gpu_son(unsigned char * d_y , unsigned char * d_u ,unsigned char * d_v , unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, int size)
{
int x = threadIdx.x + blockDim.x*blockIdx.x;
if (x >= size) return;
int rt,gt,bt;
int y, cb, cr;

y  = ((int)d_y[x]);
cb = ((int)d_u[x]) - 128;
cr = ((int)d_v[x]) - 128;

rt  = (int)( y + 1.402*cr);
gt  = (int)( y - 0.344*cb - 0.714*cr);
bt  = (int)( y + 1.772*cb);

d_r[x] = clip_rgb_gpu(rt);
d_g[x] = clip_rgb_gpu(gt);
d_b[x] = clip_rgb_gpu(bt);
}