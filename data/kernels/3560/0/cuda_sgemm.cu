#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_sgemm(float* matrix_a, float* matrix_b, float* matrix_c, size_t M, size_t K, size_t N) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

float sum = 0;
if (col < N && row < M) {
for (int k = 0; k < K; k++) {
sum +=
matrix_a[INDEX(row, k, M, K)] * matrix_b[INDEX(k, col, K, N)];
}
matrix_c[INDEX(row, col, M, N)] = sum;
}
}