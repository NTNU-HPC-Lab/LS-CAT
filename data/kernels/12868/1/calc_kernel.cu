#include "hip/hip_runtime.h"
#include "includes.h"

#define TOLERANCE 0.00001
#define TRUE 1
#define FALSE 0

long usecs();
void initialize(double **A, int rows, int cols);
int calc_serial(double **A, int rows, int cols, int iters, double tolerance);
int calc_serial_v1(double **A, int rows, int cols, int iters, double tolerance);
int calc_omp(double **A, int rows, int cols, int iters, double tolerance, int num_threads);
int calc_gpu(double **A, int rows, int cols, int iters, double tolerance);
double verify(double **A, double **B, int rows, int cols);


__global__ void calc_kernel(double* w, double* r, int rows, int cols, double tolerance) {
int row = blockIdx.x;
int col = threadIdx.x;
int idx = row*blockDim.x + col;
if (row < rows && row > 0 && col < cols) {
w[idx] = 0.2*(r[idx+1] + r[idx - 1] + r[(row-1)*blockDim.x + col] + r[(row+1)*blockDim.x + col]);
}
}