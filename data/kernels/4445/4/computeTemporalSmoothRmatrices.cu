#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
int sampleIdx = blockIdx.x;
int scanlineIdxLocal = blockIdx.y;

if (sampleIdx < numSamples)
{
int subArraySizeLocal = subArraySizes[scanlineIdxLocal * numSamples + sampleIdx];
if (subArraySizeLocal > 0)
{
int numelR = subArraySizeLocal*(subArraySizeLocal + 1) /2;
int numelRfull = subArraySize*(subArraySize + 1) /2;

int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing)) + scanlineIdxLocal * numSamples;
int lastIdx = min((int)(numSamples)-1, sampleIdx + (int)(temporalSmoothing)) + scanlineIdxLocal * numSamples;

float scaling = 1.0f;
for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
{
float finalEntry = 0.0f;
for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
{
finalEntry += Rmatrices[matrixIdx + tempIdx*numelRfull];
}
TempRmatrices[matrixIdx + (scanlineIdxLocal * numSamples + sampleIdx)*numelRfull] = finalEntry*scaling;
}
}
}
}