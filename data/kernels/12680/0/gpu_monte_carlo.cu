#include "hip/hip_runtime.h"
#include "includes.h"
// Source: http://web.mit.edu/pocky/www/cudaworkshop/MonteCarlo/Pi.cu

// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C
//
//How to run?
//===========
//
//Single precision :
//
//nvcc -O3 pi-hiprand.cu ; ./a.out <thread_num>
//
//Double precision
//
//nvcc -O3 -D DP pi-hiprand.cu ; ./a.out <thread_num>


#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256


//Help code for switching between Single Precision and Double Precision
#ifdef DP
typedef double Real;
#define PI  3.14159265358979323846  // known value of pi
#else
typedef float Real;
#define PI 3.1415926535  // known value of pi
#endif


/**
A random number generator.
Guidance from from http://stackoverflow.com/a/3067387/1281089
**/
__global__ void gpu_monte_carlo(Real *estimate, hiprandState *states, int trials) {
unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
int points_in_circle = 0;
Real x, y;

hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


for(int i = 0; i < trials; i++) {
x = hiprand_uniform (&states[tid]);
y = hiprand_uniform (&states[tid]);
points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
}
estimate[tid] = 4.0f * points_in_circle / (Real) trials; // return estimate of pi
}