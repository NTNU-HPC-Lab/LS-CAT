#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelAddConstant(int *g_a, const int b)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
g_a[idx] += b;
}