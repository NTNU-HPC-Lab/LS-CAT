#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_mi(int nbins, int nsamples, int nx, float * x_bin_scores, int pitch_x_bin_scores, float * entropies_x, int ny, float * y_bin_scores, int pitch_y_bin_scores, float * entropies_y, float * mis, int pitch_mis)
{
int
col_x = blockDim.x * blockIdx.x + threadIdx.x,
col_y = blockDim.y * blockIdx.y + threadIdx.y;

if((col_x >= nx) || (col_y >= ny))
return;

float
prob, logp, mi = 0.f,
* x_bins = x_bin_scores + col_x * pitch_x_bin_scores,
* y_bins = y_bin_scores + col_y * pitch_y_bin_scores;

// calculate joint entropy
for(int i = 0; i < nbins; i++) {
for(int j = 0; j < nbins; j++) {
prob = 0.f;
for(int k = 0; k < nsamples; k++)
prob += x_bins[k * nbins + i] * y_bins[k * nbins + j];
prob /= (float)nsamples;

if(prob <= 0.f)
logp = 0.f;
else
logp = __log2f(prob);

mi += prob * logp;
}
}

// calculate mi from entropies
mi += entropies_x[col_x] + entropies_y[col_y];
(mis + col_y * pitch_mis)[col_x] = mi;
}