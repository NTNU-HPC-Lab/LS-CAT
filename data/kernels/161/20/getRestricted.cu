#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows

// mX has order rows x cols
// vectY has length rows

__global__ void getRestricted(int countx, int county, int rows, int cols, float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim, float * mR, int mRdim, float * vectB, int vectBdim) {

int
m = blockIdx.x * THREADSPERDIM + threadIdx.x, n,
i, j, k;
float
sum, invnorm,
* X, * Y, * Q, * R, * B,
* coli, * colj,
* colQ, * colX;

if(m >= county) return;
if(m == 1) n = 0;
else n = 1;

X = mX + (m * mXdim);
// initialize the intercepts
for(i = 0; i < rows; i++)
X[i] = 1.f;

Y = vY + (m * countx + n) * vYdim;
B = vectB + m * vectBdim;
Q = mQ + m * mQdim;
R = mR + m * mRdim;

// initialize Q with X ...
for(i = 0; i < rows; i++) {
for(j = 0; j < cols; j++)
Q[i+j*rows] = X[i+j*rows];
}

// gramm-schmidt process to find Q
for(j = 0; j < cols; j++) {
colj = Q+rows*j;
for(i = 0; i < j; i++) {
coli = Q+rows*i;
sum = 0.f;
for(k = 0; k < rows; k++)
sum += coli[k] * colj[k];
for(k = 0; k < rows; k++)
colj[k] -= sum * coli[k];
}
sum = 0.f;
for(i = 0; i < rows; i++)
sum += colj[i] * colj[i];
invnorm = 1.f / sqrtf(sum);
for(i = 0; i < rows; i++)
colj[i] *= invnorm;
}
for(i = cols-1; i > -1; i--) {
colQ = Q+i*rows;
// matmult Q * X -> R
for(j = 0; j < cols; j++) {
colX = X+j*rows;
sum = 0.f;
for(k = 0; k < rows; k++)
sum += colQ[k] * colX[k];
R[i+j*cols] = sum;
}
sum = 0.f;
// compute the vector Q^t * Y -> B
for(j = 0; j < rows; j++)
sum += colQ[j] * Y[j];
// back substitution to find the x for Rx = B
for(j = cols-1; j > i; j--)
sum -= R[i+j*cols] * B[j];

B[i] = sum / R[i+i*cols];
}
}