#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "maximum_kernel_same.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *vg_a = NULL;
hipMalloc(&vg_a, XSIZE*YSIZE);
size_t pitch_a = 2;
size_t n_a = XSIZE;
const float *vg_b = NULL;
hipMalloc(&vg_b, XSIZE*YSIZE);
size_t pitch_b = 2;
size_t n_b = YSIZE;
size_t k = 1;
float *d = NULL;
hipMalloc(&d, XSIZE*YSIZE);
size_t pitch_d = 2;
float p = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
maximum_kernel_same<<<gridBlock,threadBlock>>>(vg_a,pitch_a,n_a,vg_b,pitch_b,n_b,k,d,pitch_d,p);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
maximum_kernel_same<<<gridBlock,threadBlock>>>(vg_a,pitch_a,n_a,vg_b,pitch_b,n_b,k,d,pitch_d,p);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
maximum_kernel_same<<<gridBlock,threadBlock>>>(vg_a,pitch_a,n_a,vg_b,pitch_b,n_b,k,d,pitch_d,p);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}