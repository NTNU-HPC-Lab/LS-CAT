#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows

// mX has order rows x cols
// vectY has length rows

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, int rCols, int unrCols, float * obs, int obsDim, float * rCoeffs, int rCoeffsDim, float * unrCoeffs, int unrCoeffsDim, float * rdata, int rdataDim, float * unrdata, int unrdataDim, float * dfStats) // float * dpValues)
{
int
j = blockIdx.x * THREADSPERDIM + threadIdx.x,
i = blockIdx.y * THREADSPERDIM + threadIdx.y,
idx = i*colsx + j, k, m;
float
kobs, fp = (float) p, frows = (float) rows,
rSsq, unrSsq,
rEst, unrEst,
score = 0.f,
* tObs, * tRCoeffs, * tUnrCoeffs,
* tRdata, * tUnrdata;

if((i >= colsy) || (j >= colsx)) return;
if((!diagFlag) && (i == j)) {
dfStats[idx] = 0.f;
// dpValues[idx] = 0.f;
return;
}

tObs = obs + (i*colsx+j)*obsDim;

tRCoeffs = rCoeffs + i*rCoeffsDim;
tRdata = rdata + i*rdataDim;

tUnrCoeffs = unrCoeffs + (i*colsx+j)*unrCoeffsDim;
tUnrdata = unrdata + (i*colsx+j)*unrdataDim;

rSsq = unrSsq = 0.f;
for(k = 0; k < rows; k++) {
unrEst = rEst = 0.f;
kobs = tObs[k];
for(m = 0; m < rCols; m++)
rEst += tRCoeffs[m] * tRdata[k+m*rows];
for(m = 0; m < unrCols; m++)
unrEst += tUnrCoeffs[m] * tUnrdata[k+m*rows];
rSsq   += (kobs - rEst) * (kobs - rEst);
unrSsq += (kobs - unrEst) * (kobs - unrEst);

}
score = ((rSsq - unrSsq)*(frows-2.f*fp-1.f)) / (fp*unrSsq);

if(!isfinite(score))
score = 0.f;

dfStats[idx] = score;
}