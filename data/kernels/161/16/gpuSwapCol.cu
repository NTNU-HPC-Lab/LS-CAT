#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512





// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void gpuSwapCol(int rows, float * dArray, int coli, int * dColj, int * dPivot)
{
int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;

if(rowIndex >= rows)
return;

int colj = coli + (*dColj);
float fholder;

fholder = dArray[rowIndex+coli*rows];
dArray[rowIndex+coli*rows] = dArray[rowIndex+colj*rows];
dArray[rowIndex+colj*rows] = fholder;

if((blockIdx.x == 0) && (threadIdx.x == 0)) {
int iholder = dPivot[coli];
dPivot[coli] = dPivot[colj];
dPivot[colj] = iholder;
}
}