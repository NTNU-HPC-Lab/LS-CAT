#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512





// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void UpdateHHNorms(int cols, float *dV, float *dNorms) {
// Copyright 2009, Mark Seligman at Rapid Biologics, LLC.  All rights
// reserved.

int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
if (colIndex < cols) {
float val = dV[colIndex];
dNorms[colIndex] -= val * val;
}
}