#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddToFloat( float *sum, float *out, const float *pIn )
{
(void) atomicAdd( &out[threadIdx.x], pIn[threadIdx.x] );
}