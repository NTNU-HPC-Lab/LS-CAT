#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void WriteClockValues( unsigned int *completionTimes, unsigned int *threadIDs )
{
size_t globalBlock = blockIdx.x+blockDim.x*(blockIdx.y+blockDim.y*blockIdx.z);
size_t globalThread = threadIdx.x+blockDim.x*(threadIdx.y+blockDim.y*threadIdx.z);

size_t totalBlockSize = blockDim.x*blockDim.y*blockDim.z;
size_t globalIndex = globalBlock*totalBlockSize + globalThread;

completionTimes[globalIndex] = clock();
threadIDs[globalIndex] = threadIdx.y<<4|threadIdx.x;
}