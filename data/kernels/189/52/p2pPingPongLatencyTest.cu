#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void p2pPingPongLatencyTest( void *_pLocal, void *_pRemote, uint64_t *pTimestamps, int bWait, int cIterations )
{
volatile int *pLocal = (volatile int *) _pLocal;
volatile int *pRemote = (volatile int *) _pRemote;
int pingpongValue = 0;
while ( cIterations-- ) {
*pTimestamps++ = clock64();
if ( bWait )
while ( *pLocal != pingpongValue );
bWait = 1;
pingpongValue = 1-pingpongValue;
*pRemote = pingpongValue;
}
}