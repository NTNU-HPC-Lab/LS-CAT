#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Return32( int *sum, int *out, const int *pIn )
{
extern __shared__ int s[];
s[threadIdx.x] = pIn[threadIdx.x];
__syncthreads();
(void) atomicAdd( &s[threadIdx.x], *pIn );
__syncthreads();
out[threadIdx.x] = s[threadIdx.x];
}