#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Return32( int *sum, int *out, const int *pIn )
{
out[threadIdx.x] = atomicAdd( &sum[threadIdx.x], *pIn );
}