#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddTo32( int *sum, int *out, const int *pIn )
{
(void) atomicAdd( &out[threadIdx.x], *pIn );
}