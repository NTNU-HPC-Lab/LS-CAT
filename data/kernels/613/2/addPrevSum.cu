#include "hip/hip_runtime.h"
#include "includes.h"
//Udacity HW 4
//Radix Sorting





__global__ void addPrevSum(unsigned int* blkSumsScan, unsigned int* blkScans, unsigned int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
if (i < n)
{
blkScans[i] += blkSumsScan[blockIdx.x];
}
}