#include "hip/hip_runtime.h"
#include "includes.h"
//Udacity HW 4
//Radix Sorting





__global__ void scanBlks(unsigned int *in, unsigned int *out, unsigned int n, unsigned int *blkSums)
{

extern __shared__ int blkData[];
int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
int i2 = i1 + blockDim.x;
if (i1 < n)
blkData[threadIdx.x] = in[i1];
if (i2 < n)
blkData[threadIdx.x + blockDim.x] = in[i2];
__syncthreads();


for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
{
int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1;
if (blkDataIdx < 2 * blockDim.x)
blkData[blkDataIdx] += blkData[blkDataIdx - stride];
__syncthreads();
}

for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
{
int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride;
if (blkDataIdx < 2 * blockDim.x)
blkData[blkDataIdx] += blkData[blkDataIdx - stride];
__syncthreads();
}


if (i1 < n)
out[i1] = blkData[threadIdx.x];
if (i2 < n)
out[i2] = blkData[threadIdx.x + blockDim.x];

if (blkSums != NULL && threadIdx.x == 0)
blkSums[blockIdx.x] = blkData[2 * blockDim.x - 1];

}