#include "hip/hip_runtime.h"
#include "includes.h"
//Udacity HW 4
//Radix Sorting





__global__ void scatter(unsigned int *in,unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int *d_histScan, unsigned int mask, unsigned int current_bits, unsigned int nBins)
{
if (threadIdx.x == 0)
{
unsigned int start = blockIdx.x*blockDim.x;
for (int i = start; i < min(n, start + blockDim.x) ; i++)
{
unsigned int bin = (in[i] >> current_bits) & mask;
out[d_histScan[blockIdx.x + bin*gridDim.x]] = in[i];
out_pos[d_histScan[blockIdx.x + bin*gridDim.x]] = in_pos[i];
d_histScan[blockIdx.x + bin*gridDim.x]++;
}
}
}