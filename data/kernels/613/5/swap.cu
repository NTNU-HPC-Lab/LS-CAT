#include "hip/hip_runtime.h"
#include "includes.h"
//Udacity HW 4
//Radix Sorting





__global__ void swap(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < n)
{
unsigned int temp = in[i];
in[i] = out[i];
out[i] = temp;

temp = in_pos[i];
in_pos[i] = out_pos[i];
out_pos[i] = temp;
}
}