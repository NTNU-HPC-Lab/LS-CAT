#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float* indata, float* outdata)
{
const auto i = blockIdx.x * blockDim.x + threadIdx.x;
const auto j = blockIdx.y * blockDim.y + threadIdx.y;

if (i >= Size[0] || j >= Size[1])
return;

outdata[j + i * Size[1]] = indata[j + i * Size[1]] * (Size[0] + Spacing[1]);
printf("[%u,%u] -> %.2f -> %.2f\n", i, j, indata[j + i * Size[1]], outdata[j + i * Size[1]]);
}