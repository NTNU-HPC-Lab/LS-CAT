#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void useNoTexture(float* pin, float* pout, int len)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

auto a = pin[0 + len * (i + c_size.x * (j + k * c_size.y))];
auto b = pin[1 + len * (i + c_size.x * (j + k * c_size.y))];
auto c = pin[2 + len * (i + c_size.x * (j + k * c_size.y))];

pout[i + c_size.x * (j + k * c_size.y)] = sqrtf(powf(a, 2) + powf(b, 2) + powf(c, 2));

}