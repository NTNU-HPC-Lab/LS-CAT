#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RevealNumber(int* number, unsigned int number_size)
{
printf("CudaDevice()::RevealNumber()\n");
unsigned int idx = blockDim.x * gridDim.x + threadIdx.x;
if (idx < number_size)
{
printf("Here comes: %i", number[idx]);
}
}