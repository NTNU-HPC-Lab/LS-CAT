#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void useSingleTexture(hipTextureObject_t tex, float* pout)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

float4 sample = tex3D<float4>(tex, i + 0.5, j + 0.5, k + 0.5);

pout[i + c_size.x * (j + k * c_size.y)] = sqrtf(powf(sample.x,2)+ powf(sample.y, 2)+ powf(sample.z, 2));
}