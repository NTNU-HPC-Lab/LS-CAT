#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeKernel(float* vec, int len)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

if (i >= c_size.x || j >= c_size.y || k >= c_size.z)
return;

for(auto w = 0; w < len; ++w)
{
long int id = w + len * (i + c_size.x * (j + k * c_size.y));
vec[id] = id;
}
}