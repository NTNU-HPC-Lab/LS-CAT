#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
__shared__ int tmp[TPB];

tmp[threadIdx.x] = input[threadIdx.x + blockIdx.x * blockDim.x];

__syncthreads();

if(threadIdx.x < blockDim.x / 2)
tmp[threadIdx.x] += tmp[threadIdx.x + blockDim.x / 2];

__syncthreads();

if(threadIdx.x < blockDim.x / 4)
tmp[threadIdx.x] += tmp[threadIdx.x + blockDim.x / 4];

__syncthreads();

if(threadIdx.x < blockDim.x / 8)
tmp[threadIdx.x] += tmp[threadIdx.x + blockDim.x / 8];

__syncthreads();

if(threadIdx.x == 0) {
tmp[threadIdx.x] += tmp[threadIdx.x + 1];
output[blockIdx.x] = tmp[threadIdx.x];
}
}