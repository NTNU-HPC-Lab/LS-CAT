#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
__shared__ float buffer[32];
long tx = threadIdx.x;
long bx = blockIdx.x;
long step = blockDim.x;
float *row = data + size*bx;

buffer[tx] = 0;

// get norm of axis
for (long i=tx; i<size; i+=step)
{
buffer[tx] += pow(fabs(row[i]), value);
}
// add (reduce)
for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}
// clip norms
__syncthreads();
float norm = pow(buffer[0], 1/value);
if (norm > maxnorm)
{
norm = maxnorm / (norm + 1e-7);
// renormalize
for (long i=tx; i<size; i+=step)
{
row[i] *= norm;
}
}
}