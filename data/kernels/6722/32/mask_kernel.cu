#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n && mask[i] == mask_num) x[i] = mask_num;
}