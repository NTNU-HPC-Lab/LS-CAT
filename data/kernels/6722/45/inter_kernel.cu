#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < (NX+NY)*B){
int b = i / (NX+NY);
int j = i % (NX+NY);
if (j < NX){
OUT[i] = X[b*NX + j];
} else {
OUT[i] = Y[b*NY + j - NX];
}
}
}