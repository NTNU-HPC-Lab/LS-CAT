#include "hip/hip_runtime.h"
#include "includes.h"
/*
column
A[][] = ---------------------threadIdx.y
|
|
|
|
row      |
|
|
|
|
threadIdx.x
*/



#define TILE_WIDTH 16
#define TILE_WIDTH 16

#define ar 311
#define ac_br 312
#define bc 115

using namespace std;

__global__ void mat_mul(int *d_A, int *d_B, int *d_C, int rowA, int colA, int rowB, int colB, int rowC, int colC)
{
int row, col;
row = threadIdx.x + blockIdx.x*blockDim.x;      // 0 to rowA/rowC
col = threadIdx.y + blockIdx.y*blockDim.y;      // 0 to colB/colC

if(row < rowC && col < colC)
{
for(int i = 0; i < colA; i++)               // colA = rowB
d_C[row*colC + col] += d_A[row*colA + i]*d_B[i*colB + col];
}
}// End of mat_mul function