#include "hip/hip_runtime.h"
#include "includes.h"
/*
column
A[][] = ---------------------threadIdx.y
|
|
|
|
row      |
|
|
|
|
threadIdx.x
*/



#define TILE_WIDTH 16
#define TILE_WIDTH 16

#define ar 311
#define ac_br 312
#define bc 115

using namespace std;

__global__ void mat_mul_shared(int *d_A, int *d_B, int *d_C, int rowA, int colA, int rowB, int colB, int rowC, int colC)
{
int bx = blockIdx.x,     by = blockIdx.y;
int tx = threadIdx.x,    ty = threadIdx.y;
int row = tx + bx*TILE_WIDTH;      // 0 to rowA/rowC
int col = ty + by*TILE_WIDTH;      // 0 to colB/colC

__shared__ int s_A[TILE_WIDTH][TILE_WIDTH], s_B[TILE_WIDTH][TILE_WIDTH];
int cvalue = 0;

for(int i = 0; i < (colA+TILE_WIDTH-1)/TILE_WIDTH; i++)
{
if(row < rowA && i*TILE_WIDTH+ty < colA)
s_A[tx][ty] = d_A[row*colA + i*TILE_WIDTH+ty];
else
s_A[tx][ty] = 0;

if(i*TILE_WIDTH+tx < rowB && col < colB)
s_B[tx][ty] = d_B[(i*TILE_WIDTH+tx)*colB + col];
else
s_B[tx][ty] = 0;

__syncthreads();

for(int k = 0; k < TILE_WIDTH; k++)
cvalue += s_A[tx][k]*s_B[k][ty];

__syncthreads();
}

if(row < rowC && col < colC)
d_C[row*colC + col] = cvalue;

}// End of mat_mul_shared function