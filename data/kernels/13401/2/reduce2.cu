#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Given a input tensor x with shape (N, C, D), compute x.mean(2).mean(0)
* This function is useful in batch normalization.
* Refer to https://people.maths.ox.ac.uk/gilesm/cuda/prac4/reduction.pdf.
* But the unrolling warps seems to be not working correctly for now.
*/



const int N = 256;
const int C = 1024;
const int D = 28*28;

__global__ void reduce2(const float* in, float* out) {
__shared__ float buffer[CUDA_NUM_THREADS];
const unsigned int tid = threadIdx.x;
const unsigned int c = blockIdx.x;

// load and accumulate data to buffer
buffer[tid] = 0;
for (int i = tid; i < N * D; i += blockDim.x) {
const unsigned int n = i / D;
const unsigned int d = i % D;
const unsigned int index = n * C * D + c * D + d;
buffer[tid] += in[index];
}
__syncthreads();

// do tree reduction in buffer
for (int s = blockDim.x / 2; s > 0; s >>= 1) {
if (tid < s) {
buffer[tid] += buffer[tid + s];
}
__syncthreads();
}

if (tid == 0) out[c] = buffer[0] / (N * D);
}