#include "hip/hip_runtime.h"
#include "includes.h"

//VERSION 0.8 MODIFIED 10/25/16 12:34 by Jack

// The number of threads per blocks in the kernel
// (if we define it here, then we can use its value in the kernel,
//  for example to statically declare an array in shared memory)
const int threads_per_block = 256;


// Forward function declarations
float GPU_vector_max(float *A, int N, int kernel_code, float *kernel_time, float *transfer_time);
float CPU_vector_max(float *A, int N);
float *get_random_vector(int N);
float *get_increasing_vector(int N);
float usToSec(long long time);
long long start_timer();
long long stop_timer(long long start_time, const char *name);
void die(const char *message);
void checkError();

// Main program
__global__ void vector_max_kernel(float *in, float *out, int N) {

// Determine the "flattened" block id and thread id
int block_id = blockIdx.x + gridDim.x * blockIdx.y;
int thread_id = blockDim.x * block_id + threadIdx.x;

// A single "lead" thread in each block finds the maximum value over a range of size threads_per_block
float max = 0.0;
if (threadIdx.x == 0) {

//calculate out of bounds guard
//our block size will be 256, but our vector may not be a multiple of 256!
int end = threads_per_block;
if(thread_id + threads_per_block > N)
end = N - thread_id;

//grab the lead thread's value
max = in[thread_id];

//grab values from all other threads' locations
for(int i = 1; i < end; i++) {

//if larger, replace
if(max < in[thread_id + i])
max = in[thread_id + i];
}

out[block_id] = max;

}
}