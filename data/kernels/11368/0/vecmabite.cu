#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void vecmabite( int *out, int *in, std::size_t size )
{
auto tid = threadIdx.x;
out[ tid ] = in[ 2 * tid ];
}