#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ifpairmabite( int * v, std::size_t size )
{
// Get the id of the thread ( 0 -> 99 ).
auto tid = threadIdx.x;
// Each thread fills a single element of the array.
if (!(v[tid] % 2))
v[ tid ] *= 2;
}