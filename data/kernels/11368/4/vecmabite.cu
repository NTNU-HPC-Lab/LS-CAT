#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecmabite( int *out, int *in, int threads, std::size_t size )
{
auto tid_x = threadIdx.x;
auto tid_b = blockIdx.x;
out[ tid_x  + threads * tid_b] = in[ 2 * (tid_x  + threads * tid_b) ];
}