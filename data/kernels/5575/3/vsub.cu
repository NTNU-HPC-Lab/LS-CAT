#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {





}
__global__ void vsub(const float *a, const float *b, float *c)
{
int i = blockIdx.x *blockDim.x + threadIdx.x;
c[i] = a[i] - b[i];
}