#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src) {
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}
float kernel[5][5] = {
{0.003765, 0.015019, 0.023792, 0.015019, 0.003765},
{0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
{0.023792, 0.094907, 0.150342, 0.094907, 0.023792},
{0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
{0.003765, 0.015019, 0.023792, 0.015019, 0.003765}
};
float r, g, b;
r = g = b = 0.0;
for (int i = 0; i < 5; i++) {
int tx = x + i - 2;
for (int j = 0; j < 5; j++) {
int ty = y + j - 2;
if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
r += src[(ty * width + tx) * 3] * kernel[i][j];
g += src[(ty * width + tx) * 3 + 1] * kernel[i][j];
b += src[(ty * width + tx) * 3 + 2] * kernel[i][j];
}
}
}
int idx = 3 * (y * width + x);
dst[idx] = r;
dst[idx + 1] = g;
dst[idx + 2] = b;
return;
}