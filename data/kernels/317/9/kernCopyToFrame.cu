#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernCopyToFrame(int N, uint8_t * frame, float * src) {
int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
if (idx >= N) {
return;
}
if (src[idx] < 0) {
frame[idx] = 0;
} else {
frame[idx] = (uint8_t) src[idx];
}
return;
}