#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};








__global__ void nonMaxSuppression(int N, int width, int height, unsigned char * in, unsigned char * out) {
int D = 1;
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}
int angle = in[y * width + x];
switch(angle) {
case 0:
if (out[y * width + x] < out[(y + D) * width + x] || out[y * width + x] < out[(y - D) * width + x]) {
out[y * width + x] = 0;
}
break;
case 45:
if (out[y * width + x] < out[(y + D) * width + x - D] || out[y * width + x] < out[(y - D) * width + x + D]) {
out[y * width + x] = 0;
}
break;
case 90:
if (out[y * width + x] < out[y * width + x + D] || out[y * width + x] < out[y * width + x - D]) {
out[y * width + x] = 0;
}
break;

case 135:
if (out[y * width + x] < out[(y + D) * width + x + D] || out[y * width + x] < out[(y - D) * width + x - D]) {
out[y * width + x] = 0;
}
break;
default:
break;
}
}