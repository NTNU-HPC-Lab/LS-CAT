#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};








__global__ void kernGradient(int N, int width, int height, unsigned char * in, unsigned char * gradient, unsigned char * edgeDir, float * G_x, float * G_y) {
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}
int idx, dx, dy, tx, ty;
float Gx, Gy, grad, angle;
idx = y * width + x;
Gx = Gy = 0;
for (dy = 0; dy < 3; dy++) {
ty = y + dy - 1;
for (dx = 0; dx < 3; dx++) {
tx = x + dx - 1;
if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
Gx += in[ty * width + tx] * G_x[dy * 3 + dx];
Gy += in[ty * width + tx] * G_y[dy * 3 + dx];
}
}
}
grad = sqrt(Gx * Gx + Gy * Gy);
angle = (atan2(Gx, Gy) / 3.14159f) * 180.0f;
unsigned char roundedAngle;
if (((-22.5 < angle) && (angle <= 22.5)) || ((157.5 < angle) && (angle <= -157.5))) {
roundedAngle = 0;
}
if (((-157.5 < angle) && (angle <= -112.5)) || ((22.5 < angle) && (angle <= 67.5))) {
roundedAngle = 45;
}
if (((-112.5 < angle) && (angle <= -67.5)) || ((67.5 < angle) && (angle <= 112.5))) {
roundedAngle = 90;
}
if (((-67.5 < angle) && (angle <= -22.5)) || ((112.5 < angle) && (angle <= 157.5))) {
roundedAngle = 135;
}
gradient[idx] = grad;
edgeDir[idx] = roundedAngle;
}