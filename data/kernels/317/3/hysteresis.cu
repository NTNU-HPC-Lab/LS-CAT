#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};








__global__ void hysteresis(int N, int width, int height, unsigned char * in) {
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}
int idx = y * width + x;
if (in[idx] > UPPERTHRESHOLD) {
in[idx] = 255;
} else if (in[idx] < LOWERTHRESHOLD) {
in[idx] = 0;
} else {
for (int dy = -1; dy <= 1; dy++) {
for (int dx = -1; dx <= 1; dx++) {
int nidx = (y + dy) * width + (x + dx);
if(0 <= (y + dy) && (y + dy) < height &&  0 <= (x + dx) && (x + dx) < width && in[nidx] > LOWERTHRESHOLD) {
in[nidx] = 255;
}
}
}
}
}