#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src, int kernSize, float * kernel) {
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}

float r, g, b;
r = g = b = 0.0;
for (int i = 0; i < kernSize; i++) {
int tx = x + i - kernSize/2;
for (int j = 0; j < kernSize; j++) {
int ty = y + j - kernSize/2;
if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
r += src[(ty * width + tx) * 3] * kernel[j * kernSize + i];
g += src[(ty * width + tx) * 3 + 1] * kernel[j * kernSize + i];
b += src[(ty * width + tx) * 3 + 2] * kernel[j * kernSize + i];
}
}
}
int idx = 3 * (y * width + x);
dst[idx] = r;
dst[idx + 1] = g;
dst[idx + 2] = b;
return;
}