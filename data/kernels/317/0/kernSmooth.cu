#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};








__global__ void kernSmooth(int N, int width, int height, unsigned char * in, unsigned char * out, const float * kernel, int kernSize) {
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;
if (x >= width || y >= height) {
return;
}
float c = 0.0f;
for (int i = 0; i < kernSize; i++) {
int tx = x + i - kernSize/2;
for (int j = 0; j < kernSize; j++) {
int ty = y + j - kernSize/2;
if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
c += in[ty * width + tx] * kernel[j * kernSize + i];
}
}
}
out[y * width + x] = fabs(c);
}