#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define CUDA_THREAD_NUM 1024
// must be a multiply of 2



void dotProductCPU();
__global__ void dotProductCuda(float *a, float *b, float *c) {
__shared__ float se[CUDA_THREAD_NUM];

// Calculate a.*b
se[threadIdx.x]=a[threadIdx.x+blockIdx.x*CUDA_THREAD_NUM]*b[threadIdx.x+blockIdx.x*CUDA_THREAD_NUM];
__syncthreads();

// Sum Reducto
int numActiveThreads=CUDA_THREAD_NUM/2;
while(numActiveThreads>0) {
if(threadIdx.x<numActiveThreads) {
se[threadIdx.x]=se[threadIdx.x]+se[threadIdx.x+numActiveThreads];
}
numActiveThreads=numActiveThreads/2;
__syncthreads();
}


if(threadIdx.x==0) {
c[blockIdx.x]=se[0];
//printf("BlockId: %d,  ThreadID: %d,  %f \n",blockIdx.x,threadIdx.x,c[blockIdx.x]);
}

return;
}