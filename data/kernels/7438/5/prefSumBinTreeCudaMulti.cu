#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefSumBinTreeCudaMulti(float *a, int n) {
__shared__ float shm[CUDA_THREAD_NUM];
int tid=threadIdx.x;
int bid=blockIdx.x;
int dot=2;//depth of tree

if((tid+1)%dot==0) {
shm[tid]=a[CUDA_THREAD_NUM*bid+tid]+a[CUDA_THREAD_NUM*bid+tid-1];
}
dot*=2;
__syncthreads();
while(dot<=n)  {
if((tid+1)%dot==0) {
shm[tid]=shm[tid]+shm[tid-dot/2];
}
dot*=2;
__syncthreads();
}
dot/=2;
while(dot>2) {
if((tid+1)%dot==0) {
if((tid+1)/dot!=1) {
shm[tid-dot/2]=shm[tid-dot/2]+shm[tid-dot];
}
}
dot/=2;
__syncthreads();
}

if((tid+1)%2==0) {
a[CUDA_THREAD_NUM*bid+tid]=shm[tid];
} else if(tid>0) {
a[CUDA_THREAD_NUM*bid+tid]=a[CUDA_THREAD_NUM*bid+tid]+shm[tid-1];

}

}