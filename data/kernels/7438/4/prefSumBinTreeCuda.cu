#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefSumBinTreeCuda(float *a, int n) {
__shared__ float shm[CUDA_THREAD_NUM];
int tid=threadIdx.x;
int dot=2;//depth of tree

if((tid+1)%dot==0) {
shm[tid]=a[tid]+a[tid-1];
}
dot*=2;
__syncthreads();
while(dot<=n)  {
if((tid+1)%dot==0) {
shm[tid]=shm[tid]+shm[tid-dot/2];
}
dot*=2;
__syncthreads();
}
dot/=2;
while(dot>2) {
if((tid+1)%dot==0) {
if((tid+1)/dot!=1) {
shm[tid-dot/2]=shm[tid-dot/2]+shm[tid-dot];
}
}
dot/=2;
__syncthreads();
}

if((tid+1)%2==0) {
a[tid]=shm[tid];
} else if(tid>0) {
a[tid]=a[tid]+shm[tid-1];

}

}