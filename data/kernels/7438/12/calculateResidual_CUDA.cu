#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateResidual_CUDA(float *a, float *b, float *c) {
__shared__ float se[1024];

int tid=threadIdx.x;
int bid=blockIdx.x;
int n=blockDim.x;
//   Calculate
se[tid]=fabsf(a[tid+bid*n]-b[tid+bid*n]);
__syncthreads();

//   Reducto
int numActiveThreads=n/2;
while(numActiveThreads>0) {
if(tid<numActiveThreads) {
se[tid]=se[tid]+se[tid+numActiveThreads];
}
numActiveThreads=numActiveThreads/2;
__syncthreads();
}


if(tid==0) {
atomicAdd(c,se[0]);
}
}