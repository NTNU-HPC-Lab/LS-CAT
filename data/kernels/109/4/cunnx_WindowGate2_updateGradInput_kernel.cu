#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate2_updateGradInput_kernel( float *gradInput, float *error, float* targetCentroids, const float *centroids,const float *input, const float *inputIndice, const float *outputIndice, const float* output, const float* gradOutput, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, float c, float d, float e, float lr)
{
unsigned int tx = threadIdx.x;
unsigned int k = blockIdx.x;
const float *gradOutput_k = gradOutput + outputWindowSize*k;
float *gradInput_k = gradInput + inputSize*k;


float *gradInputWindow = gradInput_k + (int)(inputIndice[k] - 1);

for (int i=tx; i<inputWindowSize; i+=blockDim.x)
{
float sum = 0;
const float *gradOutputChannel = gradOutput_k + i*windowStride;
for (int j=0; j<windowStride; j++)
sum += gradOutputChannel[j];

gradInputWindow[i] += sum;
}

}