#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cunnx_WindowGate2_updateGradInput_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *gradInput = NULL;
hipMalloc(&gradInput, XSIZE*YSIZE);
float *error = NULL;
hipMalloc(&error, XSIZE*YSIZE);
float *targetCentroids = NULL;
hipMalloc(&targetCentroids, XSIZE*YSIZE);
const float *centroids = NULL;
hipMalloc(&centroids, XSIZE*YSIZE);
const float *input = NULL;
hipMalloc(&input, XSIZE*YSIZE);
const float *inputIndice = NULL;
hipMalloc(&inputIndice, XSIZE*YSIZE);
const float *outputIndice = NULL;
hipMalloc(&outputIndice, XSIZE*YSIZE);
const float *output = NULL;
hipMalloc(&output, XSIZE*YSIZE);
const float *gradOutput = NULL;
hipMalloc(&gradOutput, XSIZE*YSIZE);
int inputSize = XSIZE*YSIZE;
int outputSize = XSIZE*YSIZE;
int inputWindowSize = XSIZE*YSIZE;
int outputWindowSize = XSIZE*YSIZE;
int windowStride = 2;
float c = 2;
float d = 1;
float e = 1;
float lr = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cunnx_WindowGate2_updateGradInput_kernel<<<gridBlock,threadBlock>>>(gradInput,error,targetCentroids,centroids,input,inputIndice,outputIndice,output,gradOutput,inputSize,outputSize,inputWindowSize,outputWindowSize,windowStride,c,d,e,lr);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cunnx_WindowGate2_updateGradInput_kernel<<<gridBlock,threadBlock>>>(gradInput,error,targetCentroids,centroids,input,inputIndice,outputIndice,output,gradOutput,inputSize,outputSize,inputWindowSize,outputWindowSize,windowStride,c,d,e,lr);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cunnx_WindowGate2_updateGradInput_kernel<<<gridBlock,threadBlock>>>(gradInput,error,targetCentroids,centroids,input,inputIndice,outputIndice,output,gradOutput,inputSize,outputSize,inputWindowSize,outputWindowSize,windowStride,c,d,e,lr);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}