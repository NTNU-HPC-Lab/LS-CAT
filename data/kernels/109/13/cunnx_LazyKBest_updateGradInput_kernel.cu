#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_LazyKBest_updateGradInput_kernel( float *gradInput, const float *indice, const float *gradOutput, int inputSize, int outputSize)
{
int tx = threadIdx.x;
int step = blockDim.x;
int k = blockIdx.x;

float *gradInput_k = gradInput + k*inputSize;
const float *gradOutput_k = gradOutput + k*outputSize;
const float *indice_k = indice + k*outputSize;

for (int i=tx; i<outputSize; i+=step)
gradInput_k[(int)(indice_k[i] - 1)] = gradOutput_k[i];
}