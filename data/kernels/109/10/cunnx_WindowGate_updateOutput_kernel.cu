#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int outputWindowSize, float a, float b, int train)
{
__shared__ float buffer[WINDOWGATE_THREADS];
unsigned int tx = threadIdx.x;
unsigned int k = blockIdx.x;
const float *input_k = input + inputSize*k;
float *output_k = output + outputWindowSize*k;

// get coordinate of centoid
buffer[tx] = 0;
for (unsigned int i=tx; i<inputSize; i+=blockDim.x)
buffer[tx] += input_k[i]*(float)(i+1);

// add (reduce)
for (unsigned int stride = WINDOWGATE_THREADS >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}

if (tx == 0)
{
float centroid = buffer[0];

// make centroid a number between 0 and 1
centroid /= (float)(inputSize);

normalizedCentroids[k] = centroid;
if ( train )
{
centroid += noise[k];
centroid = fminf(fmaxf(0,centroid),1);
}
// align centroid to output
centroid *= (float)(outputSize);

float outputIdx = centroid - 0.5*(float)outputWindowSize;

// clip indices
outputIdx = fminf(outputIdx, outputSize-outputWindowSize+1);
outputIdx = fmaxf(outputIdx, 1);

outputIdx = ceilf(outputIdx);
// align centroid to outputWindow
centroid -= (outputIdx-1);

outputIndice[k] = (int)outputIdx;
centroids[k] = centroid;
buffer[0] = centroid;
}

__syncthreads();

float centroid = buffer[0];

// gaussian blur
for (int i=tx; i<outputWindowSize; i+=blockDim.x)
{
float x = (float)(i+1)-centroid;
output_k[i] = a*expf(x*x*b);
}
}