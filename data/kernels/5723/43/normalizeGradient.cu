#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalizeGradient(float* gradient, int* activeMask, int activeSlices, int slices)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= activeSlices)
return;

int slice = activeMask[i];

float norm = gradient[6 * slices + slice];
if (norm > 0)
norm = 1.0f / sqrtf(norm);

for (int j = 0; j < 6; ++j)
gradient[j*slices + slice] *= norm;
}