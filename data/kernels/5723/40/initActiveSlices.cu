#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initActiveSlices(int* buffer, int num)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i < num)
buffer[i] = i;
}