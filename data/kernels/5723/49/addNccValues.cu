#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addNccValues(const float* prevData, float* result, int slices)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < slices)
{
float norm = prevData[3 * tid + 1] * prevData[3 * tid + 2];
float res = 0;
if (norm > 0)
res = prevData[3 * tid] / sqrtf(norm);
result[tid] += res;
}
}