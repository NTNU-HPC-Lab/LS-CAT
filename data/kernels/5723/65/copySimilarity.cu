#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copySimilarity(float* similarities, int active_patches, int patches, int* activeMask, int target, int source)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= active_patches)
return;
int patch = activeMask[i];
similarities[target*patches + patch] = similarities[source*patches + patch];
}