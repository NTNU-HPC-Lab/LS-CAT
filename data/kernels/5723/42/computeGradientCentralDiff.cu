#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeGradientCentralDiff(const float* similarities, float* gradient, int* activeMask, int activeSlices, int slices, int p)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= activeSlices)
return;
int slice = activeMask[i];

float dx = similarities[slice] - similarities[slices + slice];
gradient[p*slices + slice] = dx;
if (p == 0)
gradient[6 * slices + slice] = dx*dx;
else
gradient[6 * slices + slice] += dx*dx;
}