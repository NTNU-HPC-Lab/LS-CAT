#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addNccValues(const float* prevData, float* result, int patches)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < patches)
{
float norm = prevData[3 * tid + 1] * prevData[3 * tid + 2];
float res = 0;
if (norm > 0)
res = prevData[3 * tid] / sqrtf(norm);
result[tid] += res;
}
}