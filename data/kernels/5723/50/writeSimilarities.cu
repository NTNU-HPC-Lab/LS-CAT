#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeSimilarities(const float* nvccResults, int* activelayers, int writestep, int writenum, float* similarities, int active_slices, int slices)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < active_slices)
{
float res = nvccResults[tid];
int slice = activelayers[tid];
for (int i = 0; i < writenum; ++i)
similarities[slices*writestep*i + slice] = res;
}
}