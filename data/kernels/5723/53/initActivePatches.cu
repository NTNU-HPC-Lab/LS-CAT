#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initActivePatches(int* buffer, int num)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i < num)
buffer[i] = i;
}