#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeGradientCentralDiff(const float* similarities, float* gradient, int* activeMask, int activePatches, int patches, int p)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= activePatches)
return;
int patch = activeMask[i];

float dx = similarities[patch] - similarities[patches + patch];
gradient[p*patches + patch] = dx;
if (p == 0)
gradient[6 * patches + patch] = dx*dx;
else
gradient[6 * patches + patch] += dx*dx;
}