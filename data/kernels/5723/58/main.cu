#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "writeSimilarities.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *nvccResults = NULL;
hipMalloc(&nvccResults, XSIZE*YSIZE);
int *activelayers = NULL;
hipMalloc(&activelayers, XSIZE*YSIZE);
int writestep = 1;
int writenum = 1;
float *similarities = NULL;
hipMalloc(&similarities, XSIZE*YSIZE);
int active_patches = 1;
int patches = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
writeSimilarities<<<gridBlock,threadBlock>>>(nvccResults,activelayers,writestep,writenum,similarities,active_patches,patches);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
writeSimilarities<<<gridBlock,threadBlock>>>(nvccResults,activelayers,writestep,writenum,similarities,active_patches,patches);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
writeSimilarities<<<gridBlock,threadBlock>>>(nvccResults,activelayers,writestep,writenum,similarities,active_patches,patches);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}