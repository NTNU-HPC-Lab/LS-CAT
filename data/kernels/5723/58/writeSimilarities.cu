#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeSimilarities(const float* nvccResults, int* activelayers, int writestep, int writenum, float* similarities, int active_patches, int patches)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < active_patches)
{
float res = nvccResults[tid];
int patch = activelayers[tid];
for (int i = 0; i < writenum; ++i)
similarities[patches*writestep*i + patch] = res;
}
}