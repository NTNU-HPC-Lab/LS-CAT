#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copySimilarity(float* similarities, int active_slices, int slices, int* activeMask, int target, int source)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= active_slices)
return;
int slice = activeMask[i];
similarities[target*slices + slice] = similarities[source*slices + slice];
}