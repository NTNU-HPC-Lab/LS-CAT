#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalizeGradient(float* gradient, int* activeMask, int activePatches, int patches)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= activePatches)
return;

int patch = activeMask[i];

float norm = gradient[6 * patches + patch];
if (norm > 0)
norm = 1.0f / sqrtf(norm);

for (int j = 0; j < 6; ++j)
gradient[j*patches + patch] *= norm;
}