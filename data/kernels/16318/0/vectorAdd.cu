#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 1024


__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
int i = threadIdx.x;

if(i<n)
c[i]=a[i]+b[i];
}