#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void operate_on_block_one_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2)
{
long tid=blockIdx.x *blockDim.x + threadIdx.x;


int x_my_block = x_start - (blockIdx.x) * tileLength;
int y_my_block = y_start + (blockIdx.x) * tileLength;
//printf("%d%d%d%d\n", i,blockIdx.x + 1, x_my_block,y_my_block);


//operate_on_block (x_start, y_start, subsequence1, subsequence2);



int x_start_local = x_my_block;
int y_start_local = y_my_block;

for (int i = 1; i <= tileLength; ++i)
{
//#pragma omp parallel for
int j= (tid % tileLength) + 1;
//printf("%d-%d\n",j,i);
if (j <= i)
{
//printf("%d\n",j);
int x_my = x_start_local - (j-1)*1;
int y_my = y_start_local + (j-1)*1;
//printf("%d%d\n",x_my,y_my );
if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
{
d_Z = 1 + d_B;
}
else
{
( d_A > d_C ? d_Z = d_A : d_Z = d_C );
}
}
x_start_local = x_start_local + 1;
}

x_start_local = x_start_local - 1;
y_start_local = y_start_local + 1;

for (int i = tileLength-1; i >= 1 ; --i)
{
int j= (tid % tileLength) + 1;
if (j <= i)
{
int x_my = x_start_local - (j-1)*1;
int y_my = y_start_local + (j-1)*1;
//printf("%d%d\n",x_my,y_my );
if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
{
d_Z = 1 + d_B;
}
else
{
( d_A > d_C ? d_Z = d_A : d_Z = d_C );
}

}

y_start_local = y_start_local + 1;
}

}