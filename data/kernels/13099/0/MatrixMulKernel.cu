#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width)
{
// Calculate the row index of the d_Pelement and d_M
int Row = blockIdx.y*blockDim.y+threadIdx.y;
// Calculate the column index of d_P and d_N
int Col = blockIdx.x*blockDim.x+threadIdx.x;
if ((Row < Width) && (Col < Width))
{
float Pvalue = 0;
// each thread computes one element of the block sub-matrix
for (int k = 0; k < Width; ++k)
{
Pvalue += d_M[Row*Width+k]*d_N[k*Width+Col];
}
d_P[Row*Width+Col] = Pvalue;
}
}