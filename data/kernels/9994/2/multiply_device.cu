#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_device (double *d_a, double *d_b,int dim) {

//Declaration of required variables.
double a, b, sum;

//Retrive the thread and block specific information.
int i = threadIdx.x,j,k;

// Begine Matrix Computation.
for (j = blockIdx.x; j < dim; j += gridDim.x) {
sum = 0;
for(k=0; k<dim; k++) {
a =d_a[k *dim+i];
b =d_a[k*dim+j];
sum  = sum + a * b;
}
d_b[ i * dim + j ] = sum;
}
}