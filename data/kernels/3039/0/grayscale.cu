#include "hip/hip_runtime.h"
#include "includes.h"
/*----------------------------------------------------------------

*

* Multiprocesadores: Cuda

* Fecha: 11-Nov-2019

* Autor: A01206747 Mariana Perez
Autor: A01205559 Roberto Nuñez

* Image = 1080 x 1920
Speedup =  33.93700 ms / 0.00250 ms  = 13.5748

*--------------------------------------------------------------*/



__global__ void grayscale(unsigned char *src, unsigned char *dest, int width, int height, int nChannels) {
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;

if(y < height && x < width) {
int pos = (y * width + x) * nChannels;

unsigned char r = (float)src[pos];
unsigned char g = (float)src[pos + 1];
unsigned char b = (float)src[pos + 2];
dest[pos] = dest[pos + 1] = dest[pos + 2] = (unsigned char)0.2126 * r + 0.7152 * g + 0.0722 * b;
}
}