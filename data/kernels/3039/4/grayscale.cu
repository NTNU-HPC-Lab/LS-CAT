#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscale(unsigned char *src, unsigned char *dest, int width, int height, int nChannels) {
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;

if(y < height && x < width) {
int pos = (y * width + x) * nChannels;

float r = src[pos + 2];
float g = src[pos + 1];
float b = src[pos + 0];

dest[pos + 2] = ((0.393f * r + 0.769f * g + 0.189f * b) > 255) ? 255 : (0.393f * r + 0.769f * g + 0.189f * b);
dest[pos + 1] = ((0.349f * r + 0.686f * g + 0.168f * b) > 255) ? 255 : (0.349f * r + 0.686f * g + 0.168f * b);
dest[pos + 0] = ((0.272f * r + 0.534f * g + 0.131f * b) > 255) ? 255 : (0.272f * r + 0.534f * g + 0.131f * b);
}
}