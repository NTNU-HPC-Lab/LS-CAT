#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply(int* a, int* b, int* c, int x, int y) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int temp =  0;
if(row < x && col < x) {
for(int i = 0; i < y; i++) {
temp += a[row * y + i] * b[i * x + col];
}
}
c[row * x + col] = temp;
}