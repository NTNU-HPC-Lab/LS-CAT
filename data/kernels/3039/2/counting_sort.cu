#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void counting_sort(int* array, int *temp, int size) {
int i, j, count;
i = threadIdx.x + (blockIdx.x * blockDim.x);
if (i < size) {
count = 0;
for(j = 0; j < size; j++) {
if(array[j] < array[i]) {
count++;
} else if(array[i] == array[j] && j < i) {
count++;
}
}
temp[count] = array[i];
}
}