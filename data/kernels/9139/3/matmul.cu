#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul(double *a, double *b, double *c, int n)
{
// Get global thread ID
int Col = blockIdx.x*blockDim.x+threadIdx.x;
int Row = blockIdx.y*blockDim.y+threadIdx.y;
// Not out of bounds
if((Col<n) && (Row<n)) {// Mutliply matrices
// printf("Hello thread %d\n", threadIdx.x);
// c[Row*n + Col] = 0;
double sum = 0.0;
for(int k=0;k<n;k++) {
// c[Row*n + Col] += a[Row*n+k]*b[k*n+Col];
sum += a[Row*n+k]*b[k*n+Col];
}
c[Row*n + Col] = sum;
}
}