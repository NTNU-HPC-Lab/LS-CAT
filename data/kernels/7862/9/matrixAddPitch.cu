#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAddPitch (int *a, int *b, int*c, int pitch) {

int idx = threadIdx.x + blockIdx.x * blockDim.x;
int idy = threadIdx.y + blockIdx.y * blockDim.y;
if (idx > pitch || idy > HEIGHT) return;

c[idy * pitch + idx] = a[idy * pitch + idx] + b[idy * pitch + idx];
}