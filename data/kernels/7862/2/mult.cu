#include "hip/hip_runtime.h"
#include "includes.h"
/*
#define N 512

#define N 2048
#define THREADS_PER_BLOCK 512

*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;



__global__ void mult(int *a, int *b, int *c)
{
int pos = threadIdx.x + blockDim.x * blockIdx.x;
if (pos >= N) return;

c[pos] = a[pos] * b[pos];
}