#include "hip/hip_runtime.h"
#include "includes.h"
/*
#define N 512

#define N 2048
#define THREADS_PER_BLOCK 512

*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;



__global__ void dotProd( int *a, int *b, int *c ) {
__shared__ int temp[THREADS_PER_BLOCK];
int index = threadIdx.x + blockIdx.x * blockDim.x;

temp[threadIdx.x] = a[index] * b[index];
__syncthreads(); // Hasta que no rellenen todos los thread temp no puedo continuar...

if(threadIdx.x == 0) {
int sum = 0;
for( int i= 0; i < THREADS_PER_BLOCK; i++ ) {
sum += temp[i];
}
c[blockIdx.x] = sum;
}
}