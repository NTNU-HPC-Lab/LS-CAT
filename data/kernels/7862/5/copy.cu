#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy(int *src, int *dest)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
if (idx >= WIDTH || idy >= HEIGHT) return;

dest[idy * WIDTH + idx] = src[idy * WIDTH + idx]; // Copio tal cual con los mismos indices facil... :)
}