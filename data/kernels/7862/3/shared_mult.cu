#include "hip/hip_runtime.h"
#include "includes.h"
/*
#define N 512

#define N 2048
#define THREADS_PER_BLOCK 512

*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;



__global__ void shared_mult(int *a, int *b, int *c)
{
__shared__ int mem[THREADS_PER_BLOCK];
int pos = threadIdx.x + blockIdx.x * blockDim.x;
mem[threadIdx.x] = a[pos]  * b[pos];

__syncthreads();
c[pos] = mem[threadIdx.x];
}