#include "hip/hip_runtime.h"
#include "includes.h"
/*
#define N 512

#define N 2048
#define THREADS_PER_BLOCK 512

*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;



__global__ void dotProd( int *a, int *b, int *c ) {
__shared__ int temp[N];

temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

__syncthreads(); // Evita condición de carrera.
if( 0 == threadIdx.x ) {
int sum = 0;
for(int i = 0; i < N; i++ ) {
sum += temp[i]; //lento
}
*c = sum;
}
}