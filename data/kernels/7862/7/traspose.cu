#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void traspose(int *src, int *dest)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
if (idx >= WIDTH || idy >= HEIGHT) return;

dest[idx * HEIGHT + idy] = src[idy * WIDTH + idx]; // Cambio el valor de la matriz a la traspuesta
// con los índices de acceso a la matriz...
}