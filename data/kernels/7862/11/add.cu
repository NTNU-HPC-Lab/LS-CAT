#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int* b, int* c)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
int idy = threadIdx.y + blockIdx.y * blockDim.y;

if (idx > WIDTH || idy > HEIGHT) return;

c[idy * WIDTH + idx] = a[idy * WIDTH + idx] + b[idy * WIDTH + idx];
}