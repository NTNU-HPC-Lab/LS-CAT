#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_by_beta_kernel(float * input, float * output, float beta)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
return;

long int id = (k * c_Size.y + j) * c_Size.x + i;

output[id] = input[id] * beta;
}