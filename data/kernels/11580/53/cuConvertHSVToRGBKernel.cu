#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertHSVToRGBKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool denormalize)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int c = y*stride + x;

if (x<width && y<height)
{
// Read
float4 in = src[c];
float H = in.x;
float S = in.y;
float V = in.z;

float4 rgb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

//    float C = V*S;

//    // Denormalize
//    if (denormalize)
//      H = H*360.0f;

//    // RGB
//    H /= 60.0f;
//    float X = C*(1.0f - abs(fmod(H, 2.0f) - 1.0f));


//    if (H >= 0.0f)
//    {
//      if (H < 1.0f)
//        rgb = make_float4(C, X, 0.0f, 0.0f);
//      else if (H < 2.0f)
//        rgb = make_float4(X, C, 0.0f, 0.0f);
//      else if (H < 3.0f)
//        rgb = make_float4(0.0f, C, X, 0.0f);
//      else if (H < 4.0f)
//        rgb = make_float4(0.0f, X, C, 0.0f);
//      else if (H < 5.0f)
//        rgb = make_float4(X, 0.0f, C, 0.0f);
//      else if (H <= 6.0f)
//        rgb = make_float4(C, 0.0f, X, 0.0f);
//    }

//    float m = V-C;
//    rgb += m;

if (S == 0)
{
rgb = make_float4(V, V, V, in.w);
dst[c] = rgb;
return;
}

H /= 60.0f;
int i = floor(H);
float f = H-i;
float p = V*(1.0f - S);
float q = V*(1.0f - S*f);
float t = V*(1.0f - S*(1.0f-f));

if (i == 0)
rgb = make_float4(V, t, p, in.w);
else if (i == 1)
rgb = make_float4(q, V, p, in.w);
else if (i == 2)
rgb = make_float4(p, V, t, in.w);
else if (i == 3)
rgb = make_float4(p, q, V, in.w);
else if (i == 4)
rgb = make_float4(t, p, V, in.w);
else if (i == 5)
rgb = make_float4(V, p, q, in.w);



// Write Back
rgb.w = in.w;
dst[c] = rgb;
}
}