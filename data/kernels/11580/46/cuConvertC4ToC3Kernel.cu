#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertC4ToC3Kernel(const float4* src, size_t src_stride, float3* dst, size_t dst_stride, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int src_c = y*src_stride + x;
int dst_c = y*dst_stride + x;

if (x<width && y<height)
{
float4 val=src[src_c];
dst[dst_c] = make_float3(val.x, val.y, val.z);
}
}