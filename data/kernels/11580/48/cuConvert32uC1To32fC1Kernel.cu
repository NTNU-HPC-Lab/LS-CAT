#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvert32uC1To32fC1Kernel(const unsigned int *src, size_t src_stride, float* dst, size_t dst_stride, float mul_constant, float add_constant, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int src_c = y*src_stride + x;
int dst_c = y*dst_stride + x;

if (x<width && y<height)
{
dst[dst_c] = src[src_c] * mul_constant + add_constant;
}
}