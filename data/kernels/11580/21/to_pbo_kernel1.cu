#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to_pbo_kernel1(unsigned char* g_in, int stride_in, uchar4* g_out, int stride_out, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x<width && y<height)
{
unsigned char value = g_in[y*stride_in+x];
g_out[y*stride_out+x] = make_uchar4(value, value, value, 1);
}
}