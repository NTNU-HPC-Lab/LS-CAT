#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvert8uC3To32fC4Kernel(const unsigned char *src, size_t src_pitch, float4* dst, size_t dst_stride, float mul_constant, float add_constant, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int src_c = y*src_pitch + x*3;
int dst_c = y*dst_stride + x;

if (x<width && y<height)
{
dst[dst_c] = make_float4(src[src_c]/255.0f, src[src_c+1]/255.0f, src[src_c+2]/255.0f, 1.0f);// * mul_constant + add_constant;
}
}