#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertRGBToHSVKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool normalize)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int c = y*stride + x;

if (x<width && y<height)
{
// Read
float4 in = src[c];
float R = in.x;
float G = in.y;
float B = in.z;

float Ma = fmaxf(R, fmaxf(G, B));
float mi = fminf(R, fminf(G, B));
float C = Ma-mi;

// Hue
float H = 0.0f;
if (C != 0.0f)
{
if (Ma == R)
H = fmod((G - B)/C, 6.0f);
if (Ma == G)
H = (B - R)/C + 2.0f;
if (Ma == B)
H = (R - G)/C + 4.0f;
}

H *= 60.0f;

// Value
float V = Ma;

// Saturation
float S = 0.0f;
if (C != 0.0f)
S = C/V;

if (H < 0.0f)
H += 360.0f;

// Normalize
if (normalize)
H /= 360.0f;

// Write Back
dst[c] = make_float4(H, S, V, in.w);
}
}