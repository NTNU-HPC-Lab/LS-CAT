#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuFilterGaussZKernel_32f_C1(float* dst, float* src, const int y, const int width, const int depth, const size_t stride, const size_t slice_stride, float sigma, int kernel_size)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int z = blockIdx.y*blockDim.y + threadIdx.y;

if(x>=0 && z>= 0 && x<width && z<depth)
{
float sum = 0.0f;
int half_kernel_elements = (kernel_size - 1) / 2;

// convolve horizontally
float g0 = 1.0f / (sqrtf(2.0f * 3.141592653589793f) * sigma);
float g1 = exp(-0.5f / (sigma * sigma));
float g2 = g1 * g1;
sum = g0 * src[z*slice_stride + y*stride + x];
float sum_coeff = g0;
for (int i = 1; i <= half_kernel_elements; i++)
{
g0 *= g1;
g1 *= g2;
int cur_z = fmaxf(0, fminf(depth-1, z + i));
sum += g0 * src[cur_z*slice_stride + y*stride + x];
cur_z = fmaxf(0, fminf(depth-1, z - i));
sum += g0 * src[cur_z*slice_stride + y*stride + x];
sum_coeff += 2.0f*g0;
}
dst[z*slice_stride + y*stride + x] = sum/sum_coeff;
}
}