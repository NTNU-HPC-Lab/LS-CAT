#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertC3ToC4Kernel(const float3* src, size_t src_stride, float4* dst, size_t dst_stride, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int c_src = y*src_stride + x;
int c_dst = y*dst_stride + x;

if (x<width && y<height)
{
float3 val=src[c_src];
dst[c_dst] =  make_float4(val.x, val.y, val.z, 1.0f);
}
}