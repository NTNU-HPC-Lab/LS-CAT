#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertLABToRGBKernel(const float4* src, float4* dst, size_t stride, int width, int height)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int c = y*stride + x;

if (x<width && y<height)
{
// Read
float4 in = src[c];

float L = in.x;
float a = in.y;
float b = in.z;


// convert to XYZ
const float T1 = cbrtf(216/24389.0f);
const float fy = (L+16) / 116.0f;

float4 XYZ;
if (L > 8)
XYZ.y = fy*fy*fy;
else
XYZ.y = L / (24389/27.0f);

float fx = a/500.0f + fy;
if (fx > T1)
XYZ.x = fx*fx*fx;
else
XYZ.x = (116*fx-16) / (24389/27.0f);

float fz = fy - b/200.0f;
if (fz > T1)
XYZ.z = fz*fz*fz;
else
XYZ.z = (116*fz-16) / (24389/27.0f);


// Normalize for D65 white point
XYZ.x *= 0.950456f;
XYZ.z *= 1.088754f;

float4 rgb;
rgb.x = 3.2404542f*XYZ.x + -1.5371385f*XYZ.y + -0.4985314f*XYZ.z;
rgb.y = -0.9692660f*XYZ.x + 1.8760108f*XYZ.y + 0.0415560f*XYZ.z;
rgb.z = 0.0556434f*XYZ.x + -0.2040259f*XYZ.y + 1.0572252f*XYZ.z;
rgb.w = in.w;

dst[c] = rgb;
}
}