#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvertRGBToLABKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool isNormalized)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;
int c = y*stride + x;

if (x<width && y<height)
{
// Read
float4 in = src[c];
if (!isNormalized)
{
in.x /= 255.0f;
in.y /= 255.0f;
in.z /= 255.0f;
in.w /= 255.0f;
}

float R = in.x;
float G = in.y;
float B = in.z;


// convert to XYZ
float4 XYZ;
XYZ.x = 0.4124564f*R + 0.3575761f*G + 0.1804375f*B;
XYZ.y = 0.2126729f*R + 0.7151522f*G + 0.0721750f*B;
XYZ.z = 0.0193339f*R + 0.1191920f*G + 0.9503041f*B;

// normalize for D65 white point
XYZ.x /= 0.950456f;
XYZ.z /= 1.088754f;

float cubeRootX, cubeRootY, cubeRootZ;
const float T1 = 216/24389.0f;
const float T2 = 24389/27.0f;

if (XYZ.x > T1)
cubeRootX = cbrtf(XYZ.x);
else
cubeRootX = (T2 * XYZ.x + 16) / 116;

if (XYZ.y > T1)
cubeRootY = cbrtf(XYZ.y);
else
cubeRootY = (T2 * XYZ.y + 16) / 116;

if (XYZ.z > T1)
cubeRootZ = cbrtf(XYZ.z);
else
cubeRootZ = (T2 * XYZ.z + 16) / 116;



dst[c] = make_float4(116*cubeRootY-16, 500*(cubeRootX-cubeRootY), 200*(cubeRootY-cubeRootZ), in.w);
}
}