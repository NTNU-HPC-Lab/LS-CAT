#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_iteration(char* buffer, char* out_buffer, size_t pitch, size_t pitch_out, int width, int height)
{
const int x = blockDim.x * blockIdx.x + threadIdx.x;
const int y = blockDim.y * blockIdx.y + threadIdx.y;
if (x >= width || y >= height)
return;

int left_x = (x - 1 + width) % width;
int right_x = (x + 1) % width;
int up_y = (y - 1 + height) % height;
int down_y = (y + 1) % height;
char n_alive = buffer[up_y * pitch + left_x] + buffer[up_y * pitch + x]
+ buffer[up_y * pitch + right_x] + buffer[y * pitch + left_x]
+ buffer[y * pitch + right_x] + buffer[down_y * pitch + left_x]
+ buffer[down_y * pitch + x] + buffer[down_y * pitch + right_x];

out_buffer[y * pitch + x] =
n_alive == 3 || (buffer[y * pitch + x] && n_alive == 2);
}