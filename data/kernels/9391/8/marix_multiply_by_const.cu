#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void marix_multiply_by_const(const float *d_input, float *d_output, float val, int n) {
int index = blockDim.x * blockIdx.x + threadIdx.x;
if (index < n) {
d_output[index] = d_input[index] * val;
}
}