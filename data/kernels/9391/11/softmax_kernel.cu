#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void softmax_kernel(int64_t nrow, int64_t ncol, const float *input_data, float *output_data) {

// two dimensional thread blocks.
int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
if (y >= nrow) {
return;
}
// y_th row of input data
input_data += y * ncol;
output_data += y * ncol;
// find max for a row.
float maxval = *input_data;
for (int x = 1; x < ncol; ++x) {
maxval = max(maxval, input_data[x]);
}
// Deduct by max for a row, and raise to exp.
// in case of too large of exp, and the result will not be affected
float sum = 0;
for (int x = 0; x < ncol; ++x) {
sum += exp(input_data[x] - maxval);
}
// Compute per-row softmax.
for (int x = 0; x < ncol; ++x) {
output_data[x] = exp(input_data[x] - maxval) / sum;
}
}