#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void matrix_elementwise_add(const float *a, const float *b, float *c, int n) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < n) {
c[index] = a[index] + b[index];
}
}