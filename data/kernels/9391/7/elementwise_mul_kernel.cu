#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void elementwise_mul_kernel(const float *data_a, const float *data_b, float *output, int n) {

int index = blockDim.x * blockIdx.x + threadIdx.x;
if (index < n) {
output[index] = data_a[index] * data_b[index];
}
}