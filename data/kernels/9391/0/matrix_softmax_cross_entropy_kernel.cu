#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol, const float *input_a, const float *input_b, float *output) {
// Dynamic shared memory, size provided at kernel launch.
extern __shared__ float loss_per_row[];
// Two dimensional thread blocks.
int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x
+ threadIdx.x;
if (y >= nrow) {
return;
}
input_a += y * ncol;
input_b += y * ncol;
float maxval = *input_a;
// Find max for a row.
for (int x = 1; x < ncol; ++x) {
maxval = max(maxval, input_a[x]);
}
// Deduct by max for a row, and raise to exp.
float sum = 0;
for (int x = 0; x < ncol; ++x) {
sum += exp(input_a[x] - maxval);
}
// Compute per-row loss.
float loss = 0;
for (int x = 0; x < ncol; ++x) {
loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
}
loss_per_row[y] = loss;
__syncthreads();
// Compute reduce_mean across rows.
float mean_loss = 0;
// Use a single thread to reduce mean across rows.
if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
for (int i = 0; i < nrow; ++i) {
mean_loss += loss_per_row[i];
}
mean_loss /= nrow;
output[0] = mean_loss;
}
}