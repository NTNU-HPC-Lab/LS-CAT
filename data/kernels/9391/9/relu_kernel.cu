#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void relu_kernel(const float *input, float *output, int n) {
int index = blockDim.x * blockIdx.x + threadIdx.x;
if (index < n) {
float element = input[index];
if (element <= 0) {
output[index] = 0;
} else {
output[index] = element;
}
}
}