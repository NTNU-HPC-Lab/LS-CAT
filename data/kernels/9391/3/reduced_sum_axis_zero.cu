#include "hip/hip_runtime.h"
#include "includes.h"

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)




__global__ void reduced_sum_axis_zero(const float *input_data, float *output_data, int input_n, int output_n) {
int idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < output_n) {
output_data[idx] = 0.0;
for (int i = 0; i < input_n / output_n; i++) {
output_data[idx] += input_data[i * output_n + idx];
}
}
}