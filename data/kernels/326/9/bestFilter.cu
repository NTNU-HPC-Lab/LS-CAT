#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  bestFilter(const double *Params, const float *data, const float *mu, const float *lam, const float *nu, float *xbest, float *err, int *ftype){

int tid, tid0, i, bid, NT, Nfilt, ibest = 0;
float Th,  Cf, Ci, xb, Cbest = 0.0f, epu, cdiff;

tid 		= threadIdx.x;
bid 		= blockIdx.x;
NT 		= (int) Params[0];
Nfilt 	= (int) Params[1];
Th 		= (float) Params[2];
epu       = (float) Params[8];

tid0 = tid + bid * Nthreads;
if (tid0<NT-1 & tid0>0){
for (i=0; i<Nfilt;i++){
Ci = data[tid0 + NT * i] + mu[i] * lam[i];
Cf = Ci * Ci / (lam[i] + 1.0f) - lam[i]*mu[i]*mu[i];

// add the shift component
cdiff = data[tid0+1 + NT * i] - data[tid0-1 + NT * i];
Cf = Cf + cdiff * cdiff / (epu + nu[i]);
if (Cf > Cbest){
Cbest 	= Cf;
xb      = Ci  - mu[i] * lam[i]; /// (lam[i] + 1);
ibest 	= i;
}
}
if (Cbest > Th*Th){
err[tid0] 	= Cbest;
xbest[tid0] 	= xb;
ftype[tid0] 	= ibest;
}
}
}