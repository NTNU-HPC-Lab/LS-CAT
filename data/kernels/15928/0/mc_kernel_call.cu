#include "hip/hip_runtime.h"
#include "includes.h"



// wrapper pour une option d'achat
__global__ void mc_kernel_call(float * d_s, float T, float K, float S0, float sigma, float mu, float r, float dt, float * d_normals, unsigned N_STEPS, unsigned N_PATHS)
{
const unsigned tid = threadIdx.x; // id du thread dans le bloc
const unsigned bid = blockIdx.x; // id du bloc
const unsigned bsz = blockDim.x; // taille du bloc

int s_idx = tid + bid * bsz;
int n_idx = tid + bid * bsz;
float s_curr = S0;

if (s_idx < N_PATHS) {
int n = 0;
do {
s_curr = s_curr + mu*s_curr*dt + sigma*s_curr*d_normals[n_idx];
n_idx++;
n++;
} while (n < N_STEPS);
double payoff = (s_curr>K ? s_curr - K : 0.0);
__syncthreads(); // on attend que tous les threads aient fini avant de passer à la prochaine simulation
d_s[s_idx] = exp(-r*T) * payoff;
}
}