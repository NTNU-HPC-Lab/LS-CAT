#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initTempNodeArray( const int hitNum, const int allowableGap, int* tempNodeArray_score, int* tempNodeArray_vertical, int* tempNodeArray_horizontal, int* tempNodeArray_matchNum) {
const int bIdx = gridDim.x * blockIdx.y + blockIdx.x;
const int idx  = blockDim.x * bIdx + threadIdx.x;
const int halfTempNodeWidth = allowableGap + MARGIN;
const int tempNodeWidth     = 1 + 2 * halfTempNodeWidth;
if(idx < hitNum * tempNodeWidth) {
const int bandIdx = idx / hitNum;
if(bandIdx < halfTempNodeWidth) {
tempNodeArray_score     [idx] = -30000;
tempNodeArray_vertical  [idx] = -30000;
tempNodeArray_horizontal[idx] = -30000;
tempNodeArray_matchNum  [idx] = -30000;
} else if(bandIdx == halfTempNodeWidth) {
tempNodeArray_score     [idx] = 0;
tempNodeArray_vertical  [idx] = GAP_OPEN_POINT;
tempNodeArray_horizontal[idx] = GAP_OPEN_POINT;
tempNodeArray_matchNum  [idx] = 0;
} else {
const int i = bandIdx - halfTempNodeWidth;
const int tempScore = i * GAP_POINT + GAP_OPEN_POINT;
tempNodeArray_score     [idx] = tempScore;
tempNodeArray_vertical  [idx] = tempScore + GAP_OPEN_POINT;
tempNodeArray_horizontal[idx] = tempScore;
tempNodeArray_matchNum  [idx] = 0;
}
}
}