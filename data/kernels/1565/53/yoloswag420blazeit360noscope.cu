#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}