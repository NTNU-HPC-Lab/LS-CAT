#include "hip/hip_runtime.h"
#include "includes.h"
/// Copyright (C) 2016 Giuseppe Bilotta <giuseppe.bilotta@gmail.com>
/// License: GPLv3


#define restrict __restrict__

typedef unsigned int uint;
typedef unsigned int hashKey;
typedef ushort4 particleinfo;

__global__ void initParticles( particleinfo * restrict infoArray, hashKey * restrict hashArray, uint * restrict idxArray, uint numParticles)
{
uint idx = threadIdx.x + blockIdx.x*blockDim.x;

if (idx > numParticles)
return;

idxArray[idx] = idx;

particleinfo info;
info.x = idx % 4;
info.y = 0;
info.z = (ushort)(idx & 0xffff);
info.w = (ushort)(idx >> 16);

infoArray[idx] = info;

hashArray[idx] = idx/17 + (idx % (idx & 17));
}