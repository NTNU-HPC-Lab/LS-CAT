#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _fill_gradBias(float *gradBias, const float *gradOutput, float scale, int batch_n, int output_n, int output_h, int output_w) {
gradOutput += blockIdx.x*output_h*output_w;
__shared__ float shGrad[128]; // 32*4
float g = .0f;
int oz,oxy;
for (oz = threadIdx.y; oz < batch_n; oz += 4) {
const float *out = gradOutput + oz*output_n*output_h*output_w;
for (oxy = threadIdx.x; oxy < output_h*output_w; oxy += 32) {
g += out[oxy];
}
}
shGrad[threadIdx.y*blockDim.x+threadIdx.x] = g;
__syncthreads();

// reduce
if (threadIdx.x == 0) {
g = .0f;
for (oxy = 0; oxy < 128; ++oxy)
g += shGrad[oxy];
gradBias[blockIdx.x] = scale*g;
}
}