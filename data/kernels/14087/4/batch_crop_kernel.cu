#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void batch_crop_kernel(float* input, const int nCropRows, const int nCropCols, const int iH, const int iW, const int nPlanes){
const int plane = blockIdx.x;
if (plane >= nPlanes)
return;

input += plane * iH * iW;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int tz = threadIdx.z;

// top
if (tz == 0) {
input[ty*iW + tx] = 0;
}
// bottom
if (tz == 1) {
input[(iH-ty-1)*iW + tx] = 0;
}
// left
if (tz == 2) {
input[tx*iW+ty] = 0;
}
// right
if (tz == 3) {
input[tx*iW + (iW-ty-1)] = 0;
}

/*
if (ty < iH && (ty > iH-nCropRows-1 || ty < nCropRows)) {
input[ty*iW + tx] = 0;
}
if (tx < iW && (tx > iW-nCropCols-1 || tx < nCropCols)) {
input[ty*iW + tx] = 0;
}
*/
}