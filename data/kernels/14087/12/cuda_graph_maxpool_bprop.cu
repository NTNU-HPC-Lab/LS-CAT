#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_graph_maxpool_bprop(float* gradInput, const float *gradOutput, const float* indices, const int nClusters, const int dim, const int nClustersPerThread) {

extern __shared__ float shared_mem[];
float* gradOutput_data = (float*)shared_mem;
float* indices_data = (float*)&gradOutput_data[nClusters];

const int tidx = threadIdx.x;
gradInput += blockIdx.x * dim;
gradOutput += blockIdx.x * nClusters;
indices += blockIdx.x * nClusters;
__syncthreads();
for (int i = 0; i < nClustersPerThread; ++i) {
int idx = tidx + i*blockDim.x;
if (idx < nClusters) {
gradOutput_data[idx] = gradOutput[idx];
indices_data[idx] = indices[idx];
}
}
__syncthreads();

//ouch...
if (tidx == 1) {
for (int i = 0; i < nClusters; ++i) {
gradInput[(int)indices_data[i]-1] += gradOutput[i];
}
}
//gradInput[(int)indices_data[tidx]-1] = gradOutput[tidx];
}