#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modulus_updateGradInput_kernel(float* input, float* output, float* gradInput, float* gradOutput, int n) {
const int i = threadIdx.x + blockIdx.x*blockDim.x;
if (i >= n)
return;
const float eps = 0.0001;
const float c = gradOutput[i]/max(output[i],eps);
gradInput[2*i] = input[2*i]*c;
gradInput[2*i+1] = input[2*i+1]*c;
}