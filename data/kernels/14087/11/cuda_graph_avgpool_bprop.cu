#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_graph_avgpool_bprop(float* gradInput, const float *gradOutput, const float* clusters, const int nClusters, const int poolsize, const int dim, const int nClustersPerThread) {

extern __shared__ float shared_mem[];
float* gradOutput_data = (float*)shared_mem;

const int tidx = threadIdx.x;
gradInput += blockIdx.x * dim;
gradOutput += blockIdx.x * nClusters;
__syncthreads();
for (int i = 0; i < nClustersPerThread; ++i) {
int idx = tidx + i*blockDim.x;
if (idx < nClusters) {
gradOutput_data[idx] = gradOutput[idx];
}
}
__syncthreads();


if (tidx < poolsize) {
for (int i = 0; i < nClusters; ++i) {
gradInput[(int)(clusters[i*poolsize+tidx]-1)] += gradOutput[i]/poolsize;
}
}

/*
for (int j = 0; j < poolsize; ++j) {
gradInput[(int)(clusters[tidx*poolsize+j]-1)] += gradOutput[tidx]/poolsize;
__syncthreads();
}
*/
__syncthreads();

/*
//ouch...
if (tidx == 1) {
for (int i = 0; i < nClusters; ++i) {
//    int idx = tidx + i*blockDim.x;
for (int j = 0; j < poolsize; ++j) {
gradInput[(int)(clusters[i*poolsize+j]-1)] += gradOutput[i]/poolsize;
}
}
}
*/




/*
for (int i = 0; i < nClustersPerThread; ++i) {
int idx = tidx + i*blockDim.x;
if (idx < nClusters) {
for (int j = 0; j < poolsize; ++j) {
gradInput[(int)clusters[idx*poolsize+j]] += gradOutput_data[idx]/poolsize;
}
}
}
*/
}