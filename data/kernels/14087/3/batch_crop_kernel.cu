#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void batch_crop_kernel(float* input, const int nCropRows, const int nCropCols, const int iH, const int iW, const int nPlanes){
const int plane = blockIdx.x;
if (plane >= nPlanes)
return;

input += plane * iH * iW;
const int tx = threadIdx.x;
const int ty = threadIdx.y;

if (ty < iH && (ty > iH-nCropRows-1 || ty < nCropRows)) {
input[ty*iW + tx] = 0;
}
if (tx < iW && (tx > iW-nCropCols-1 || tx < nCropCols)) {
input[ty*iW + tx] = 0;
}
}