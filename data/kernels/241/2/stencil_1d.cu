#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out)
{
// __shared__ keyword to declare variables in shared block memory
__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
int lindex = threadIdx.x + RADIUS;

temp[lindex] = in[gindex];
if (threadIdx.x < RADIUS)
{
temp[lindex - RADIUS] = in[gindex - RADIUS];
temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
}

__syncthreads();

// Apply the stencil
int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
result += temp[lindex + offset];

// Store the result
out[gindex-RADIUS] = result;
}