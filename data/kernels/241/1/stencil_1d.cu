#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out)
{
// blockDim is 3-dimensional vector storing block grid dimensions

// index of a thread across all threads + RADIUS
int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;

int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
result += in[gindex + offset];

// Store the result
out[gindex - RADIUS] = result;
}