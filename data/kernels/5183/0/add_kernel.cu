#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void add_kernel(Para* para, int size)
{
int tid = threadIdx.x;

if (tid < size)
{
para->d_c[tid] = para->d_a[tid] + para->d_b[tid];
}
}