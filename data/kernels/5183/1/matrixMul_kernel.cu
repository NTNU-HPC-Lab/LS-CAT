#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul_kernel(float * A, float * B, float * C, int N)
{
int ROW = blockIdx.y * blockDim.y + threadIdx.y;
int COL = blockIdx.x * blockDim.x + threadIdx.x;

float tmpSum = 0;

if (ROW < N && COL < N)
{
// each thread computes one elem of the block sub-matrix
for (int i = 0; i < N; i++)
{
tmpSum += A[ROW * N + i] * B[i * N + COL];
}
}
C[ROW * N + COL] = tmpSum;
}