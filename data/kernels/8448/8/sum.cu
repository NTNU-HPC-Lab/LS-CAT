#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(float *a, float *b, float *c) {
int index = blockDim.x * blockIdx.x + threadIdx.x;
if (index < N) {
c[index] = a[index] + b[index];
}
}