#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *cudaA, float *kernel, float *cudaResult)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
int gid = idy * N + idx;

__shared__ float blockData[BLOCK_SIZE + 2 * BLUR_SIZE][BLOCK_SIZE + 2 * BLUR_SIZE][3];

int x = idx - BLUR_SIZE;
int y = idy - BLUR_SIZE;

if(x >= 0 && y >= 0)
for(int k = 0; k < 3; k++)
blockData[threadIdx.x][threadIdx.y][k] = cudaA[(gid - BLUR_SIZE - BLUR_SIZE * N)*3 + k];
else
for(int k = 0; k < 3; k++)
blockData[threadIdx.x][threadIdx.y][k] = 0;

x = idx + BLUR_SIZE;
y = idy - BLUR_SIZE;

if(x < N && y >= 0)
for(int k = 0; k < 3; k++)
blockData[threadIdx.y][threadIdx.x + 2 * BLUR_SIZE][k] = cudaA[(gid + BLUR_SIZE - BLUR_SIZE * N)*3 + k];
else
for(int k = 0; k < 3; k++)
blockData[threadIdx.y][threadIdx.x + 2 * BLUR_SIZE][k] = 0;

x = idx - BLUR_SIZE;
y = idy + BLUR_SIZE;

if(x >= 0 && y < N)
for(int k = 0; k < 3; k++)
blockData[threadIdx.y + 2 * BLUR_SIZE][threadIdx.x][k] = cudaA[(gid - BLUR_SIZE + BLUR_SIZE * N)*3 + k];
else
for(int k = 0; k < 3; k++)
blockData[threadIdx.y + 2 * BLUR_SIZE][threadIdx.x][k] = 0;

x = idx + BLUR_SIZE;
y = idy + BLUR_SIZE;

if(x < N && y < N)
for(int k = 0; k < 3; k++)
blockData[threadIdx.y + 2 * BLUR_SIZE][threadIdx.x + 2 * BLUR_SIZE][k] = cudaA[(gid + BLUR_SIZE + BLUR_SIZE * N)*3 + k];
else
for(int k = 0; k < 3; k++)
blockData[threadIdx.y + 2 * BLUR_SIZE][threadIdx.x + 2 * BLUR_SIZE][k] = 0;

__syncthreads();
for(int k = 0; k < 3; k++)
{
for(int i = -BLUR_SIZE; i <= BLUR_SIZE; i++)
for(int j = -BLUR_SIZE; j <= BLUR_SIZE; j++)
{
cudaResult[gid * 3 + k] += blockData[threadIdx.y + BLUR_SIZE + i][threadIdx.x + BLUR_SIZE + j][k] * kernel[(BLUR_SIZE - i) * (2 * BLUR_SIZE + 1) + (BLUR_SIZE - j)];
}
}
}