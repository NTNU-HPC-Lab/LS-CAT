#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ void print(float *result)
{
for(int k = 0; k < 3; k++)
{
for(int i = 0; i < N; i++)
{
for(int j = 0; j < N; j++)
printf("%f ",result[(i*N + j)*3 + k]);
printf("\n");
}
printf("\n");
}
}
__global__ void histogram_creation(int *A, int *hist, int no_of_threads) {

int global_x = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ int local_hist[N+1];

for(int i = threadIdx.x; i<=N; i = i + (blockDim.x ) ){
local_hist[i] = 0;
}
__syncthreads();

for(int i = global_x; i <= M; i = i + (blockDim.x * no_of_threads)) {
atomicAdd(&local_hist[A[i]],1);
}
__syncthreads();

for(int i = threadIdx.x ; i <= N; i = i + (blockDim.x) ) {
atomicAdd(&hist[i],local_hist[i]);
printf("%d histogram_local %d \n",local_hist[i],i);
}
__syncthreads();

}