#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tiled_matrix_multiplication(int *A, int *B, int *C) {

__shared__ int As[TILE_WIDTH][TILE_WIDTH];
__shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

int res = 0;

for(int i = 0; i < M/TILE_WIDTH; i++) {
As[ty][tx] = A[row * M + (i*TILE_WIDTH + tx)];
Bs[ty][tx] = B[(i*TILE_WIDTH + ty)* M + col];

__syncthreads();

for(int j = 0; j < TILE_WIDTH; j++) {
res += As[ty][j] + Bs[j][tx];
}

__syncthreads();
}

C[row * M + col] = res;

}