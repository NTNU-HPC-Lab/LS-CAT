#include "hip/hip_runtime.h"
#include "includes.h"
/*
Vector addition with a single thread for each addition
*/



/*
Vector addition with thread mapping and thread accessing its neighbor parallely
*/

//slower than simpler


/*
Matrix Matrix multiplication with a single thread for each row
*/


/*
Matrix Matrix multiplication with a single thread for each result element
*/


/*
Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/


__global__ void simple_addition(int *a, int *b,int *c,int len)
{
int tid=threadIdx.x +blockIdx.x*blockDim.x ;
//while (tid<len)
c[tid]=a[tid]+b[tid];
//printf("I am block: %d with tid: %d Result: %d \n",blockIdx.x,tid,c[tid]);

}