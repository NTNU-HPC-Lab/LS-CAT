#include "hip/hip_runtime.h"
#include "includes.h"
/*
Vector addition with a single thread for each addition
*/



/*
Vector addition with thread mapping and thread accessing its neighbor parallely
*/

//slower than simpler


/*
Matrix Matrix multiplication with a single thread for each row
*/


/*
Matrix Matrix multiplication with a single thread for each result element
*/


/*
Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/


__global__ void matrix_matrix_mul_old(int *a, int *b, int *c, int n_row, int n_col, int n_comm)

{
int tid= threadIdx.x + blockIdx.x * blockDim.x;
int temp=0;
while(tid<n_row)
{
for (int k=0;k<n_col;k++)
{
temp=0;
for(int j=0;j<n_comm;j++)
{
temp+= a[n_comm*tid+j]* b[j*n_col+k];
}
c[tid*n_col+k]=temp;
}
tid+=blockDim.x * gridDim.x;

}
}