#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define CHUNK_SIZE 16

// flag if the prng has been seeded
int randNotSeeded = 1;

// tests the gpu merge sort
__device__ void insertionSort(int *array, int a, int b)
{
int current;
for (int i = a + 1; i < b; i++)
{
current = array[i];
for (int j = i - 1; j >= a - 1; j--)
{
if (j == a - 1 || current > array[j])
{
array[j + 1] = current;
break;
}
else
{
array[j + 1] = array[j];
}
}
}
}
__global__ void gpu_sort(int *d_array, int size, int chunkSize)
{
// Figure out left and right for this thread
int a = (threadIdx.x + blockDim.x * blockIdx.x) * chunkSize;
if (a >= size) return;

int b = a + chunkSize;
if (b > size) b = size;

insertionSort(d_array, a, b);
}