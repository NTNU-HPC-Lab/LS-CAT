#include "hip/hip_runtime.h"
#include "includes.h"
/*CUDA 2-D Matrix Multiplication*/


#define TILE_WIDTH 2
#define WIDTH  100


// main routine
__global__ void MatrixMul( float *A_d , float *B_d , float *C_d)
{
// calculate thread id
unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

C_d[row*WIDTH+col] = 0;
for (int k = 0 ; k<WIDTH ; k++ )
{
C_d[row*WIDTH + col]+= A_d[row * WIDTH + k ] * B_d[ k * WIDTH + col] ;
}
}