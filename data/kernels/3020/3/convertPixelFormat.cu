#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPixelFormat(uint8_t* inputBgra, uint8_t* outputYuv, int numPixels){
int stride = gridDim.x * blockDim.x;
int idx = threadIdx.x + blockIdx.x * blockDim.x;
short3 yuv16;
char3 yuv8;

while(idx<=numPixels){
if(idx<numPixels){
yuv16.x = 66*inputBgra[idx*4+2] + 129*inputBgra[idx*4+1] + 25*inputBgra[idx*4];
yuv16.y = -38*inputBgra[idx*4+2] + -74*inputBgra[idx*4+1] + 112*inputBgra[idx*4];
yuv16.z = 112*inputBgra[idx*4+2] + -94*inputBgra[idx*4+1] + -18*inputBgra[idx*4];

yuv8.x = (yuv16.x>>8)+16;
yuv8.y = (yuv16.y>>8)+128;
yuv8.z = (yuv16.z>>8)+128;

*(reinterpret_cast<char3*>(&outputYuv[idx*3])) = yuv8;
}
idx += stride;
}
}