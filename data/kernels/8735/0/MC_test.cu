#include "hip/hip_runtime.h"
#include "includes.h"
/**************************************
***************************************
* Code Can be compiled using --> nvcc kernel5.cu -lcurand if the cuRand lib is the envirement PATH
* else use nvcc kernel5.cu -L</path/to/the/lib> -lcurand
***************************************
**************************************/



__global__ void MC_test(unsigned int seed,hiprandState *states,unsigned int numsim,unsigned int *results)
{
extern __shared__ int sdata[];
int i;
int nthreads = gridDim.x * blockDim.x;
unsigned int innerpoint=0;
int tx=threadIdx.x;
int idx = blockIdx.x * blockDim.x + tx;
hiprandState *state =states + idx;
float x,y,l2norm2;
sdata[tx]=0;
__syncthreads();
hiprand_init(seed, tx, 0, state);
__syncthreads();
for(i=tx;i<numsim;i+=nthreads){
x = hiprand_uniform(state);
y = hiprand_uniform(state);
l2norm2 = x * x + y * y;
if (l2norm2 < static_cast<float>(1))
{
innerpoint++;;
}
}
__syncthreads();
sdata[tx]=innerpoint;
__syncthreads();
//-------reduction
for (unsigned int s=blockDim.x/2;s>0;s>>=1){
if(tx < s){
sdata[tx]=sdata[tx]+sdata[tx+s];
}
}
//-----------------
__syncthreads();
if(tx==0){
results[blockIdx.x]=sdata[0];
}

}