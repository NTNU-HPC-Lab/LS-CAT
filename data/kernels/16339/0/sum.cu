#include "hip/hip_runtime.h"
#include "includes.h"

#define N 18



__global__ void sum(double *a, double *b, double *c) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}