#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy(float a, float* x, float* y) {
// RUN: sh -c "test `grep -c -F 'y[hipThreadIdx_x] = a * x[hipThreadIdx_x];' %t` -eq 2"
y[threadIdx.x] = a * x[threadIdx.x];
}