#include "hip/hip_runtime.h"
#include "includes.h"
// richu shaji abraham richursa
using namespace std;
__device__ int function(int value , int bit ,int bitset)
{
if(bitset == 1 )
{
if((value & bit)  != 0)
{
return 1;
}
else
return 0;
}
else
{
if((value & bit) == 0)
{
return 1;
}
else
{
return 0;
}
}
}
__global__ void predicateDevice(int *d_array , int *d_predicateArrry , int d_numberOfElements,int bit,int bitset)
{
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < d_numberOfElements)
{

d_predicateArrry[index] = function(d_array[index],bit,bitset);
}
}