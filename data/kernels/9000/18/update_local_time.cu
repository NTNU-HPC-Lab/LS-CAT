#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_local_time(int *next, double *local_time, double GTIME){

unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
int who = next[gtid];

if(who < 0)
return;

local_time[who] = GTIME;

}