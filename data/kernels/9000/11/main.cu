#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "dev_get_gravity_at_point.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float eps2 = 1;
float *eps = NULL;
hipMalloc(&eps, XSIZE*YSIZE);
float *xh = NULL;
hipMalloc(&xh, XSIZE*YSIZE);
float *yh = NULL;
hipMalloc(&yh, XSIZE*YSIZE);
float *zh = NULL;
hipMalloc(&zh, XSIZE*YSIZE);
float *xt = NULL;
hipMalloc(&xt, XSIZE*YSIZE);
float *yt = NULL;
hipMalloc(&yt, XSIZE*YSIZE);
float *zt = NULL;
hipMalloc(&zt, XSIZE*YSIZE);
float *ax = NULL;
hipMalloc(&ax, XSIZE*YSIZE);
float *ay = NULL;
hipMalloc(&ay, XSIZE*YSIZE);
float *az = NULL;
hipMalloc(&az, XSIZE*YSIZE);
int n = XSIZE*YSIZE;
float *field_m = NULL;
hipMalloc(&field_m, XSIZE*YSIZE);
float *fxh = NULL;
hipMalloc(&fxh, XSIZE*YSIZE);
float *fyh = NULL;
hipMalloc(&fyh, XSIZE*YSIZE);
float *fzh = NULL;
hipMalloc(&fzh, XSIZE*YSIZE);
float *fxt = NULL;
hipMalloc(&fxt, XSIZE*YSIZE);
float *fyt = NULL;
hipMalloc(&fyt, XSIZE*YSIZE);
float *fzt = NULL;
hipMalloc(&fzt, XSIZE*YSIZE);
int n_field = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
dev_get_gravity_at_point<<<gridBlock,threadBlock>>>(eps2,eps,xh,yh,zh,xt,yt,zt,ax,ay,az,n,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
dev_get_gravity_at_point<<<gridBlock,threadBlock>>>(eps2,eps,xh,yh,zh,xt,yt,zt,ax,ay,az,n,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
dev_get_gravity_at_point<<<gridBlock,threadBlock>>>(eps2,eps,xh,yh,zh,xt,yt,zt,ax,ay,az,n,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}