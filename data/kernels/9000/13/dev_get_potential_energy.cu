#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dev_get_potential_energy( float *partial_results, float eps2, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
extern __shared__ float thread_results[];
unsigned int i, j;
float dx, dy, dz, r, dr2, potential_energy = 0;
for (j=threadIdx.x + blockIdx.x*blockDim.x; j < n_field; j += blockDim.x*gridDim.x){
for (i=0; i<j; i++){
dx = (fxh[i] - fxh[j]) + (fxt[i] - fxt[j]);
dy = (fyh[i] - fyh[j]) + (fyt[i] - fyt[j]);
dz = (fzh[i] - fzh[j]) + (fzt[i] - fzt[j]);
dr2 = dx*dx + dy*dy + dz*dz;
r = sqrt(eps2 + dr2);
potential_energy -= field_m[i]*field_m[j] / r;
}
}

// Reduce results from all threads within this block
thread_results[threadIdx.x] = potential_energy;
__syncthreads();
for (i = blockDim.x/2; i>0; i>>=1) {
if (threadIdx.x < i) {
thread_results[threadIdx.x] += thread_results[threadIdx.x + i];
}
__syncthreads();
}
if (threadIdx.x == 0) {
partial_results[blockIdx.x] = thread_results[0];
}
}