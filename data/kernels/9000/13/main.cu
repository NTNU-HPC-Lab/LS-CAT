#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "dev_get_potential_energy.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *partial_results = NULL;
hipMalloc(&partial_results, XSIZE*YSIZE);
float eps2 = 1;
float *field_m = NULL;
hipMalloc(&field_m, XSIZE*YSIZE);
float *fxh = NULL;
hipMalloc(&fxh, XSIZE*YSIZE);
float *fyh = NULL;
hipMalloc(&fyh, XSIZE*YSIZE);
float *fzh = NULL;
hipMalloc(&fzh, XSIZE*YSIZE);
float *fxt = NULL;
hipMalloc(&fxt, XSIZE*YSIZE);
float *fyt = NULL;
hipMalloc(&fyt, XSIZE*YSIZE);
float *fzt = NULL;
hipMalloc(&fzt, XSIZE*YSIZE);
int n_field = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
dev_get_potential_energy<<<gridBlock,threadBlock>>>(partial_results,eps2,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
dev_get_potential_energy<<<gridBlock,threadBlock>>>(partial_results,eps2,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
dev_get_potential_energy<<<gridBlock,threadBlock>>>(partial_results,eps2,field_m,fxh,fyh,fzh,fxt,fyt,fzt,n_field);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}