#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Reconstruct.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *nex = NULL;
hipMalloc(&nex, XSIZE*YSIZE);
unsigned long nextsize = 1;
double4 *pc = NULL;
hipMalloc(&pc, XSIZE*YSIZE);
double4 *vc = NULL;
hipMalloc(&vc, XSIZE*YSIZE);
double4 *a3 = NULL;
hipMalloc(&a3, XSIZE*YSIZE);
double4 *a = NULL;
hipMalloc(&a, XSIZE*YSIZE);
double4 *a1 = NULL;
hipMalloc(&a1, XSIZE*YSIZE);
double4 *a2 = NULL;
hipMalloc(&a2, XSIZE*YSIZE);
double4 *pva3 = NULL;
hipMalloc(&pva3, XSIZE*YSIZE);
double4 *aaa = NULL;
hipMalloc(&aaa, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Reconstruct<<<gridBlock,threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Reconstruct<<<gridBlock,threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Reconstruct<<<gridBlock,threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}