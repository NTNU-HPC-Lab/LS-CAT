#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_partial(double4 *a, double4 *b, unsigned int nextsize){

unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

if(i >= nextsize)
return;

extern __shared__ double4 shaccelerations[];
double4 *shacc = (double4*) shaccelerations;
double4 myacc;

myacc = b[i];
shacc[threadIdx.x] = a[i];

myacc.x += shacc[threadIdx.x].x;
myacc.y += shacc[threadIdx.x].y;
myacc.z += shacc[threadIdx.x].z;

b[i] = myacc;

}