#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Corrector_gpu(double GTIME, double *local_time, double *step, int *next, unsigned long nextsize, double4 *pos_CH, double4 *vel_CH, double4 *a_tot_D, double4 *a1_tot_D, double4 *a2_tot_D, double4 *a_H0, double4 *a3_H, double ETA6, double ETA4, double DTMAX, double DTMIN, unsigned int N){

unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;

double dt;
int who = next[gtid];
int who1 = gtid + nextsize;
int who2 = who1 + nextsize;

if(gtid >= nextsize )
return;

a_H0[gtid].w = a_H0[gtid].x * a_H0[gtid].x +
a_H0[gtid].y * a_H0[gtid].y +
a_H0[gtid].z * a_H0[gtid].z ;

a_H0[who1].w = a_H0[who1].x * a_H0[who1].x +
a_H0[who1].y * a_H0[who1].y +
a_H0[who1].z * a_H0[who1].z ;

a_H0[who2].w =  a_H0[who2].x * a_H0[who2].x +
a_H0[who2].y * a_H0[who2].y +
a_H0[who2].z * a_H0[who2].z ;

double h = GTIME-local_time[who];
local_time[who] = GTIME;

double h1 = 0.5*h;
double h2 = h1*h1;
double h3 = 0.75/(h1*h1*h1);
double h4 = 1.5/(h2*h2);
double h5 = 7.5/(h2*h2*h1);

double Amin = a_H0[gtid].x - a_tot_D[who].x;
double Aplu = a_H0[gtid].x + a_tot_D[who].x;
double Jmin = h1 * (a_H0[who1].x - a1_tot_D[who].x);
double Jplu = h1 * (a_H0[who1].x + a1_tot_D[who].x);
double Smin = h1 * h1 * (a_H0[who2].x - a2_tot_D[who].x);
double Splu = h1 * h1 * (a_H0[who2].x + a2_tot_D[who].x);

double over= 1.0/15.0;

pos_CH[who].x = pos_CH[who].x + h1*vel_CH[who].x - 0.4*h2*Amin + over*h2*Jplu;
vel_CH[who].x = vel_CH[who].x + h1*Aplu          - 0.4*h1*Jmin + over*h1*Splu;
pos_CH[who].x += h1*vel_CH[who].x;

a3_H[who].x = h3*(-5.0*Amin + 5.0*Jplu - Smin);
double a4halfx = h4*(-Jmin + Splu);
double a5halfx = h5*(3.0*Amin - 3.0*Jplu + Smin);
a3_H[who].x += h1*a4halfx + 0.5*h2*a5halfx;
a4halfx += h1*a5halfx;

Amin = a_H0[gtid].y - a_tot_D[who].y;
Aplu = a_H0[gtid].y + a_tot_D[who].y;
Jmin = h1 * (a_H0[who1].y - a1_tot_D[who].y);
Jplu = h1 * (a_H0[who1].y + a1_tot_D[who].y);
Smin = h1 * h1 * (a_H0[who2].y - a2_tot_D[who].y);
Splu = h1 * h1 * (a_H0[who2].y + a2_tot_D[who].y);

pos_CH[who].y = pos_CH[who].y + h1*vel_CH[who].y - 0.4*h2*Amin + over*h2*Jplu;
vel_CH[who].y = vel_CH[who].y + h1*Aplu          - 0.4*h1*Jmin + over*h1*Splu;
pos_CH[who].y += h1*vel_CH[who].y;

a3_H[who].y = h3*(-5.0*Amin + 5.0*Jplu - Smin);
double a4halfy = h4*(-Jmin + Splu);
double a5halfy = h5*(3.0*Amin - 3.0*Jplu + Smin);
a3_H[who].y += h1*a4halfy + 0.5*h2*a5halfy;
a4halfy += h1*a5halfy;

Amin = a_H0[gtid].z - a_tot_D[who].z;
Aplu = a_H0[gtid].z + a_tot_D[who].z;
Jmin = h1 * (a_H0[who1].z - a1_tot_D[who].z);
Jplu = h1 * (a_H0[who1].z + a1_tot_D[who].z);
Smin = h1 * h1 * (a_H0[who2].z - a2_tot_D[who].z);
Splu = h1 * h1 * (a_H0[who2].z + a2_tot_D[who].z);

pos_CH[who].z = pos_CH[who].z + h1*vel_CH[who].z - 0.4*h2*Amin + over*h2*Jplu;
vel_CH[who].z = vel_CH[who].z + h1*Aplu          - 0.4*h1*Jmin + over*h1*Splu;
pos_CH[who].z += h1*vel_CH[who].z;

a3_H[who].z = h3*(-5.0*Amin + 5.0*Jplu - Smin);
double a4halfz = h4*(-Jmin + Splu);
double a5halfz = h5*(3.0*Amin - 3.0*Jplu + Smin);
a3_H[who].z += h1*a4halfz + 0.5*h2*a5halfz;
a4halfz += h1*a5halfz;

a3_H[who].w = sqrt(a3_H[who].x*a3_H[who].x + a3_H[who].y*a3_H[who].y + a3_H[who].z*a3_H[who].z);
double a4mod = sqrt(a4halfx*a4halfx + a4halfy*a4halfy + a4halfz*a4halfz);
double a5mod = sqrt(a5halfx*a5halfx + a5halfy*a5halfy + a5halfz*a5halfz);

double    dt6 = (sqrt(a_H0[gtid].w*a_H0[who2].w) + a_H0[who1].w) / (a5mod*a3_H[who].w + a4mod*a4mod);
dt6 = ETA6 * pow(dt6,1.0/6.0);

double stp = h;
double overh3 = 1.0/(stp*stp*stp);
double overh2 = 1.0/(stp*stp);

double a2dx = overh2 * (-6.0 * (a_tot_D[who].x - a_H0[gtid].x) -
stp * (4.0 * a_H0[who1].x + 2.0 * a1_tot_D[who].x));
double a2dy = overh2 * (-6.0 * (a_tot_D[who].y - a_H0[gtid].y) -
stp * (4.0 * a_H0[who1].y + 2.0 * a1_tot_D[who].y));
double a2dz = overh2 * (-6.0 * (a_tot_D[who].z - a_H0[gtid].z) -
stp * (4.0 * a_H0[who1].z + 2.0 * a1_tot_D[who].z));

double a3dx = overh3 * (12.0 * (a_tot_D[who].x - a_H0[gtid].x) +
6.0 * stp * (a_H0[who1].x + a1_tot_D[who].x));
double a3dy = overh3 * (12.0 * (a_tot_D[who].y - a_H0[gtid].y) +
6.0 * stp * (a_H0[who1].y + a1_tot_D[who].y));
double a3dz = overh3 * (12.0 * (a_tot_D[who].z - a_H0[gtid].z) +
6.0 * stp * (a_H0[who1].z + a1_tot_D[who].z));

a2dx += h*a3dx;
a2dy += h*a3dy;
a2dx += h*a3dz;

a_H0[who2].w =  a2dx*a2dx + a2dy*a2dy + a2dz*a2dz;
a3_H[who].w = a3dx*a3dx + a3dy*a3dy + a3dz*a3dz;

double dt4 = sqrt(ETA4*(sqrt(a_H0[gtid].w*a_H0[who2].w) + a_H0[who1].w) / (sqrt(a_H0[who1].w*a3_H[who].w) + a_H0[who2].w));

dt = 0.5*dt4+0.5*dt6;

double rest = GTIME / (2.0 * step[who]);
rest = (double)((int)(rest)) - rest;

//	return;
//	pos_CH[who].x = step[who];
//	return;

if(dt > 2.0*step[who] && rest == 0.0 && 2.0*step[who] <= DTMAX)
step[who] *= 2.0;
else if (dt < 0.5*step[who])
step[who] *= 0.25;
else if (dt < step[who])
step[who]*=0.5;

if(step[who] < DTMIN)
step[who] = DTMIN;

a_tot_D[who] = a_H0[gtid];
a1_tot_D[who] = a_H0[who1];
a2_tot_D[who] = a_H0[who2];

}