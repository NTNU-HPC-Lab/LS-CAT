#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Corrector_gpu.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double GTIME = 1;
double *local_time = NULL;
hipMalloc(&local_time, XSIZE*YSIZE);
double *step = NULL;
hipMalloc(&step, XSIZE*YSIZE);
int *next = NULL;
hipMalloc(&next, XSIZE*YSIZE);
unsigned long nextsize = 1;
double4 *pos_CH = NULL;
hipMalloc(&pos_CH, XSIZE*YSIZE);
double4 *vel_CH = NULL;
hipMalloc(&vel_CH, XSIZE*YSIZE);
double4 *a_tot_D = NULL;
hipMalloc(&a_tot_D, XSIZE*YSIZE);
double4 *a1_tot_D = NULL;
hipMalloc(&a1_tot_D, XSIZE*YSIZE);
double4 *a2_tot_D = NULL;
hipMalloc(&a2_tot_D, XSIZE*YSIZE);
double4 *a_H0 = NULL;
hipMalloc(&a_H0, XSIZE*YSIZE);
double4 *a3_H = NULL;
hipMalloc(&a3_H, XSIZE*YSIZE);
double ETA6 = 1;
double ETA4 = 1;
double DTMAX = 1;
double DTMIN = 1;
unsigned int N = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Corrector_gpu<<<gridBlock,threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Corrector_gpu<<<gridBlock,threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Corrector_gpu<<<gridBlock,threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}