#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dev_get_potential_at_point( float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *phi, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
float dx, dy, dz, r, dr2, eps2_total;
for (int tid=threadIdx.x + blockIdx.x*blockDim.x; tid < n; tid += blockDim.x*gridDim.x){
eps2_total = eps2 + eps[tid]*eps[tid];
phi[tid] = 0;
for (int i=0; i < n_field; i++){
dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
dr2 = dx*dx + dy*dy + dz*dz;
if (dr2 > 0) {
r = sqrt(eps2_total + dr2);
phi[tid] -= field_m[i] / r;
}
}
}
}