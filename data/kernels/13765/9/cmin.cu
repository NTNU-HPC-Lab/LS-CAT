#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cmin(float *d_in, float *min, int len)
{
extern __shared__ float smin[];

unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x  + threadIdx.x;

smin[tid] = d_in[i]<d_in[i+len] ? d_in[i] : d_in[i+len];

__syncthreads();
if(blockDim.x > 512 && tid<512) {if(smin[tid] > smin[tid+512]) smin[tid] = smin[tid+512];}  __syncthreads();
if(blockDim.x > 256 && tid<256) {if(smin[tid] > smin[tid+256]) smin[tid] = smin[tid+256];}  __syncthreads();
if(blockDim.x > 128 && tid<128) {if(smin[tid] > smin[tid+128]) smin[tid] = smin[tid+128];}  __syncthreads();
if(blockDim.x > 64 && tid<64) {if(smin[tid] > smin[tid+64]) smin[tid] = smin[tid+64];}  __syncthreads();
if(tid<32) {
if(blockDim.x > 32 && smin[tid] > smin[tid+32]) smin[tid] = smin[tid+32];
if(blockDim.x > 16 && smin[tid] > smin[tid+16]) smin[tid] = smin[tid+16];
if(blockDim.x > 8 && smin[tid] > smin[tid+8]) smin[tid] = smin[tid+8];
if(blockDim.x > 4 && smin[tid] > smin[tid+4]) smin[tid] = smin[tid+4];
if(blockDim.x > 2 && smin[tid] > smin[tid+2]) smin[tid] = smin[tid+2];
if(smin[tid] > smin[tid+1]) smin[tid] = smin[tid+1];
__syncthreads();
}

if(tid == 0 )
{
min[blockIdx.x] = smin[0];
}
}