#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  blelloch_no_padding(unsigned int* d_in_array, const size_t numBins)
/*

\Params:
* d_in_array - input array of histogram values in each bin. Gets converted
to cdf by the end of the function.
* numBins - number of bins in the histogram (Must be < 2*MAX_THREADS_PER_BLOCK)
*/
{

int thid = threadIdx.x;

extern __shared__ float temp_array[];

temp_array[thid] = d_in_array[thid];
temp_array[thid + numBins/2] = d_in_array[thid + numBins/2];

__syncthreads();

// Part 1: Up Sweep, reduction
int stride = 1;
for (int d = numBins>>1; d > 0; d>>=1) {

if (thid < d) {
int neighbor = stride*(2*thid+1) - 1;
int index = stride*(2*thid+2) - 1;

temp_array[index] += temp_array[neighbor];
}
stride *=2;
__syncthreads();
}
// Now set last element to identity:
if (thid == 0)  temp_array[numBins-1] = 0;

// Part 2: Down sweep
for (int d=1; d<numBins; d *= 2) {
stride >>= 1;
__syncthreads();

if(thid < d) {
int neighbor = stride*(2*thid+1) - 1;
int index = stride*(2*thid+2) - 1;

float t = temp_array[neighbor];
temp_array[neighbor] = temp_array[index];
temp_array[index] += t;
}
}

__syncthreads();

d_in_array[thid] = temp_array[thid];
d_in_array[thid + numBins/2] = temp_array[thid + numBins/2];

}