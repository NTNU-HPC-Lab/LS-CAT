#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_histogram(unsigned int* bins, const float* dIn, const int binNumber, const float lumMin, const float lumMax, const int size) {

unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i > size)
return;

float range = lumMax - lumMin;
int bin = ((dIn[i] - lumMin) / range) * binNumber;

atomicAdd(&bins[bin], 1);
}