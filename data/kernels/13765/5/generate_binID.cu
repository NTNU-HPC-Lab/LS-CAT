#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_binID(const float* dIn, int* out, const int binNumber, const float lumMin, const float lumMax, const int size) {
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i>size)
{
return;
}

float range = lumMax - lumMin;
int bin = ((dIn[i] - lumMin) / range) * binNumber;

out[i] = bin;
}