#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cmax(float *d_in, float *max, int len)
{
extern __shared__ float smax[];

unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x  + threadIdx.x;

smax[tid] = d_in[i]>d_in[i+len] ? d_in[i] : d_in[i+len];

__syncthreads();
if(blockDim.x > 512 && tid<512) {if(smax[tid] < smax[tid+512]) smax[tid] = smax[tid+512];}  __syncthreads();
if(blockDim.x > 256 && tid<256) {if(smax[tid] < smax[tid+256]) smax[tid] = smax[tid+256];}  __syncthreads();
if(blockDim.x > 128 && tid<128) {if(smax[tid] < smax[tid+128]) smax[tid] = smax[tid+128];}  __syncthreads();
if(blockDim.x > 64 && tid<64) {if(smax[tid] < smax[tid+64]) smax[tid] = smax[tid+64];}  __syncthreads();
if(tid<32) {
if(blockDim.x > 32 && smax[tid] < smax[tid+32]) smax[tid] = smax[tid+32];
if(blockDim.x > 16 && smax[tid] < smax[tid+16]) smax[tid] = smax[tid+16];
if(blockDim.x > 8 && smax[tid] < smax[tid+8]) smax[tid] = smax[tid+8];
if(blockDim.x > 4 && smax[tid] < smax[tid+4]) smax[tid] = smax[tid+4];
if(blockDim.x > 2 && smax[tid] < smax[tid+2]) smax[tid] = smax[tid+2];
if(smax[tid] < smax[tid+1]) smax[tid] = smax[tid+1];
__syncthreads();
}
if(tid == 0 )
{
max[blockIdx.x] = smax[0];
}
}