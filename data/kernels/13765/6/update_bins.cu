#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_bins(unsigned int* bins, int* in_binID, int binNumber, const int size){
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x;
int nt = blockDim.x * blockDim.y;

__shared__ unsigned int temp[1024];
temp[tid] = 0;
__syncthreads();

for(int x=tid; x<size; x+=nt){
if(in_binID[x] == i){;
temp[tid]++;
}
if(in_binID[x] > i){
break;
}
}
__syncthreads();

if(tid == 0){
for(int x = 0; x<binNumber;x++){
bins[i] += temp[x];
}
}


}