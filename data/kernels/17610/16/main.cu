#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "sortAtomsGenCellListsAlt.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned int natoms = 1;
const float4 *xyzr_d = NULL;
hipMalloc(&xyzr_d, XSIZE*YSIZE);
const float4 *color_d = NULL;
hipMalloc(&color_d, XSIZE*YSIZE);
const unsigned int *atomIndex_d = NULL;
hipMalloc(&atomIndex_d, XSIZE*YSIZE);
unsigned int *sorted_atomIndex_d = NULL;
hipMalloc(&sorted_atomIndex_d, XSIZE*YSIZE);
const unsigned int *atomHash_d = NULL;
hipMalloc(&atomHash_d, XSIZE*YSIZE);
float4 *sorted_xyzr_d = NULL;
hipMalloc(&sorted_xyzr_d, XSIZE*YSIZE);
float4 *sorted_color_d = NULL;
hipMalloc(&sorted_color_d, XSIZE*YSIZE);
uint2 *cellStartEnd_d = NULL;
hipMalloc(&cellStartEnd_d, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sortAtomsGenCellListsAlt<<<gridBlock,threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
sortAtomsGenCellListsAlt<<<gridBlock,threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
sortAtomsGenCellListsAlt<<<gridBlock,threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}