#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortAtomsGenCellListsAlt(unsigned int natoms, const float4 *xyzr_d, const float4 *color_d, const unsigned int *atomIndex_d, unsigned int *sorted_atomIndex_d, const unsigned int *atomHash_d, float4 *sorted_xyzr_d, float4 *sorted_color_d, uint2 *cellStartEnd_d) {
extern __shared__ unsigned int hash_s[]; // blockSize + 1 elements
unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
unsigned int hash;

// do nothing if current index exceeds the number of atoms
if (index < natoms) {
hash = atomHash_d[index];
hash_s[threadIdx.x+1] = hash; // use smem to avoid redundant loads
if (index > 0 && threadIdx.x == 0) {
// first thread in block must load neighbor particle hash
hash_s[0] = atomHash_d[index-1];
}
}

__syncthreads();

if (index < natoms) {
// Since atoms are sorted, if this atom has a different cell
// than its predecessor, it is the first atom in its cell, and
// it's index marks the end of the previous cell.
if (index == 0 || hash != hash_s[threadIdx.x]) {
cellStartEnd_d[hash].x = index; // set start
if (index > 0)
cellStartEnd_d[hash_s[threadIdx.x]].y = index; // set end
}

if (index == natoms - 1) {
cellStartEnd_d[hash].y = index + 1; // set end
}

// Reorder atoms according to sorted indices
unsigned int sortedIndex = atomIndex_d[index];
sorted_atomIndex_d[sortedIndex] = index;
float4 pos = xyzr_d[sortedIndex];
sorted_xyzr_d[index] = pos;

// Reorder colors according to sorted indices, if provided
if (color_d != NULL) {
float4 col = color_d[sortedIndex];
sorted_color_d[index] = col;
}
}
}