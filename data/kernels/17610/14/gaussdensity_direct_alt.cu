#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void gaussdensity_direct_alt(int natoms, const float4 *xyzr, float gridspacing, unsigned int z, float *densitygrid) {
unsigned int xindex  = (blockIdx.x * blockDim.x) * DUNROLLX + threadIdx.x;
unsigned int yindex  = (blockIdx.y * blockDim.y) + threadIdx.y;
unsigned int zindex  = (blockIdx.z * blockDim.z) + threadIdx.z;
unsigned int outaddr =
((gridDim.x * blockDim.x) * DUNROLLX) * (gridDim.y * blockDim.y) * zindex +
((gridDim.x * blockDim.x) * DUNROLLX) * yindex + xindex;
zindex += z;

float coorx = gridspacing * xindex;
float coory = gridspacing * yindex;
float coorz = gridspacing * zindex;

float densityvalx1=0.0f;
float densityvalx2=0.0f;
#if DUNROLLX >= 4
float densityvalx3=0.0f;
float densityvalx4=0.0f;
#endif
#if DUNROLLX >= 8
float densityvalx5=0.0f;
float densityvalx6=0.0f;
float densityvalx7=0.0f;
float densityvalx8=0.0f;
#endif

float gridspacing_coalesce = gridspacing * DBLOCKSZX;

int atomid;
for (atomid=0; atomid<natoms; atomid++) {
float4 atom = xyzr[atomid];
float dy = coory - atom.y;
float dz = coorz - atom.z;
float dyz2 = dy*dy + dz*dz;

float dx1 = coorx - atom.x;
float r21 = (dx1*dx1 + dyz2) * atom.w;
densityvalx1 += exp2f(-r21);

float dx2 = dx1 + gridspacing_coalesce;
float r22 = (dx2*dx2 + dyz2) * atom.w;
densityvalx2 += exp2f(-r22);

#if DUNROLLX >= 4
float dx3 = dx2 + gridspacing_coalesce;
float r23 = (dx3*dx3 + dyz2) * atom.w;
densityvalx3 += exp2f(-r23);

float dx4 = dx3 + gridspacing_coalesce;
float r24 = (dx4*dx4 + dyz2) * atom.w;
densityvalx4 += exp2f(-r24);
#endif
#if DUNROLLX >= 8
float dx5 = dx4 + gridspacing_coalesce;
float r25 = (dx5*dx5 + dyz2) * atom.w;
densityvalx5 += exp2f(-r25);

float dx6 = dx5 + gridspacing_coalesce;
float r26 = (dx6*dx6 + dyz2) * atom.w;
densityvalx6 += exp2f(-r26);

float dx7 = dx6 + gridspacing_coalesce;
float r27 = (dx7*dx7 + dyz2) * atom.w;
densityvalx7 += exp2f(-r27);

float dx8 = dx7 + gridspacing_coalesce;
float r28 = (dx8*dx8 + dyz2) * atom.w;
densityvalx8 += exp2f(-r28);
#endif
}

densitygrid[outaddr             ] += densityvalx1;
densitygrid[outaddr+1*DBLOCKSZX] += densityvalx2;
#if DUNROLLX >= 4
densitygrid[outaddr+2*DBLOCKSZX] += densityvalx3;
densitygrid[outaddr+3*DBLOCKSZX] += densityvalx4;
#endif
#if DUNROLLX >= 8
densitygrid[outaddr+4*DBLOCKSZX] += densityvalx5;
densitygrid[outaddr+5*DBLOCKSZX] += densityvalx6;
densitygrid[outaddr+6*DBLOCKSZX] += densityvalx7;
densitygrid[outaddr+7*DBLOCKSZX] += densityvalx8;
#endif
}