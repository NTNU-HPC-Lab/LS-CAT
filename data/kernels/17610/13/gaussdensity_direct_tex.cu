#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void gaussdensity_direct_tex(int natoms, const float4 *xyzr, const float4 *colors, float gridspacing, unsigned int z, float *densitygrid, float3 *voltexmap, float invisovalue) {
unsigned int xindex  = (blockIdx.x * blockDim.x) * DUNROLLX + threadIdx.x;
unsigned int yindex  = (blockIdx.y * blockDim.y) + threadIdx.y;
unsigned int zindex  = (blockIdx.z * blockDim.z) + threadIdx.z;
unsigned int outaddr =
((gridDim.x * blockDim.x) * DUNROLLX) * (gridDim.y * blockDim.y) * zindex +
((gridDim.x * blockDim.x) * DUNROLLX) * yindex + xindex;
zindex += z;

float coorx = gridspacing * xindex;
float coory = gridspacing * yindex;
float coorz = gridspacing * zindex;

float densityvalx1=0.0f;
float densityvalx2=0.0f;
float3 densitycolx1;
densitycolx1=make_float3(0.0f, 0.0f, 0.0f);
float3 densitycolx2=densitycolx1;

#if DUNROLLX >= 4
float densityvalx3=0.0f;
float densityvalx4=0.0f;
float3 densitycolx3=densitycolx1;
float3 densitycolx4=densitycolx1;
#endif
#if DUNROLLX >= 8
float densityvalx5=0.0f;
float densityvalx6=0.0f;
float densityvalx7=0.0f;
float densityvalx8=0.0f;

float3 densitycolx5=densitycolx1;
float3 densitycolx6=densitycolx1;
float3 densitycolx7=densitycolx1;
float3 densitycolx8=densitycolx1;
#endif

float gridspacing_coalesce = gridspacing * DBLOCKSZX;

int atomid;
for (atomid=0; atomid<natoms; atomid++) {
float4 atom = xyzr[atomid];
float4 color = colors[atomid];

float dy = coory - atom.y;
float dz = coorz - atom.z;
float dyz2 = dy*dy + dz*dz;

float dx1 = coorx - atom.x;
float r21 = (dx1*dx1 + dyz2) * atom.w;
float tmp1 = exp2f(-r21);
densityvalx1 += tmp1;
tmp1 *= invisovalue;
densitycolx1.x += tmp1 * color.x;
densitycolx1.y += tmp1 * color.y;
densitycolx1.z += tmp1 * color.z;

float dx2 = dx1 + gridspacing_coalesce;
float r22 = (dx2*dx2 + dyz2) * atom.w;
float tmp2 = exp2f(-r22);
densityvalx2 += tmp2;
tmp2 *= invisovalue;
densitycolx2.x += tmp2 * color.x;
densitycolx2.y += tmp2 * color.y;
densitycolx2.z += tmp2 * color.z;

#if DUNROLLX >= 4
float dx3 = dx2 + gridspacing_coalesce;
float r23 = (dx3*dx3 + dyz2) * atom.w;
float tmp3 = exp2f(-r23);
densityvalx3 += tmp3;
tmp3 *= invisovalue;
densitycolx3.x += tmp3 * color.x;
densitycolx3.y += tmp3 * color.y;
densitycolx3.z += tmp3 * color.z;

float dx4 = dx3 + gridspacing_coalesce;
float r24 = (dx4*dx4 + dyz2) * atom.w;
float tmp4 = exp2f(-r24);
densityvalx4 += tmp4;
tmp4 *= invisovalue;
densitycolx4.x += tmp4 * color.x;
densitycolx4.y += tmp4 * color.y;
densitycolx4.z += tmp4 * color.z;
#endif
#if DUNROLLX >= 8
float dx5 = dx4 + gridspacing_coalesce;
float r25 = (dx5*dx5 + dyz2) * atom.w;
float tmp5 = exp2f(-r25);
densityvalx5 += tmp5;
tmp5 *= invisovalue;
densitycolx5.x += tmp5 * color.x;
densitycolx5.y += tmp5 * color.y;
densitycolx5.z += tmp5 * color.z;

float dx6 = dx5 + gridspacing_coalesce;
float r26 = (dx6*dx6 + dyz2) * atom.w;
float tmp6 = exp2f(-r26);
densityvalx6 += tmp6;
tmp6 *= invisovalue;
densitycolx6.x += tmp6 * color.x;
densitycolx6.y += tmp6 * color.y;
densitycolx6.z += tmp6 * color.z;

float dx7 = dx6 + gridspacing_coalesce;
float r27 = (dx7*dx7 + dyz2) * atom.w;
float tmp7 = exp2f(-r27);
densityvalx7 += tmp7;
tmp7 *= invisovalue;
densitycolx7.x += tmp7 * color.x;
densitycolx7.y += tmp7 * color.y;
densitycolx7.z += tmp7 * color.z;

float dx8 = dx7 + gridspacing_coalesce;
float r28 = (dx8*dx8 + dyz2) * atom.w;
float tmp8 = exp2f(-r28);
densityvalx8 += tmp8;
tmp8 *= invisovalue;
densitycolx8.x += tmp8 * color.x;
densitycolx8.y += tmp8 * color.y;
densitycolx8.z += tmp8 * color.z;
#endif
}

densitygrid[outaddr             ] += densityvalx1;
voltexmap[outaddr             ].x += densitycolx1.x;
voltexmap[outaddr             ].y += densitycolx1.y;
voltexmap[outaddr             ].z += densitycolx1.z;

densitygrid[outaddr+1*DBLOCKSZX] += densityvalx2;
voltexmap[outaddr+1*DBLOCKSZX].x += densitycolx2.x;
voltexmap[outaddr+1*DBLOCKSZX].y += densitycolx2.y;
voltexmap[outaddr+1*DBLOCKSZX].z += densitycolx2.z;

#if DUNROLLX >= 4
densitygrid[outaddr+2*DBLOCKSZX] += densityvalx3;
voltexmap[outaddr+2*DBLOCKSZX].x += densitycolx3.x;
voltexmap[outaddr+2*DBLOCKSZX].y += densitycolx3.y;
voltexmap[outaddr+2*DBLOCKSZX].z += densitycolx3.z;

densitygrid[outaddr+3*DBLOCKSZX] += densityvalx4;
voltexmap[outaddr+3*DBLOCKSZX].x += densitycolx4.x;
voltexmap[outaddr+3*DBLOCKSZX].y += densitycolx4.y;
voltexmap[outaddr+3*DBLOCKSZX].z += densitycolx4.z;
#endif
#if DUNROLLX >= 8
densitygrid[outaddr+4*DBLOCKSZX] += densityvalx5;
voltexmap[outaddr+4*DBLOCKSZX].x += densitycolx5.x;
voltexmap[outaddr+4*DBLOCKSZX].y += densitycolx5.y;
voltexmap[outaddr+4*DBLOCKSZX].z += densitycolx5.z;

densitygrid[outaddr+5*DBLOCKSZX] += densityvalx6;
voltexmap[outaddr+5*DBLOCKSZX].x += densitycolx6.x;
voltexmap[outaddr+5*DBLOCKSZX].y += densitycolx6.y;
voltexmap[outaddr+5*DBLOCKSZX].z += densitycolx6.z;

densitygrid[outaddr+6*DBLOCKSZX] += densityvalx7;
voltexmap[outaddr+6*DBLOCKSZX].x += densitycolx7.x;
voltexmap[outaddr+6*DBLOCKSZX].y += densitycolx7.y;
voltexmap[outaddr+6*DBLOCKSZX].z += densitycolx7.z;

densitygrid[outaddr+7*DBLOCKSZX] += densityvalx8;
voltexmap[outaddr+7*DBLOCKSZX].x += densitycolx8.x;
voltexmap[outaddr+7*DBLOCKSZX].y += densitycolx8.y;
voltexmap[outaddr+7*DBLOCKSZX].z += densitycolx8.z;
#endif
}