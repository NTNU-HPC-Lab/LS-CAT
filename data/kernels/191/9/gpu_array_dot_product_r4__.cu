#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_dot_product_r4__(size_t tsize, const float *arr1, const float *arr2, volatile float *dprod)
{
extern __shared__ float dprs_r4[]; //volume = blockDim.x
size_t l;
int i,j;
float dpr;
dpr=0.0f; for(l=blockIdx.x*blockDim.x+threadIdx.x;l<tsize;l+=gridDim.x*blockDim.x){dpr+=arr1[l]*arr2[l];}
dprs_r4[threadIdx.x]=dpr;
__syncthreads();
i=1; while(i < blockDim.x){j=threadIdx.x*(i*2); if(j+i < blockDim.x) dprs_r4[j]+=dprs_r4[j+i]; i*=2;}
__syncthreads();
if(threadIdx.x == 0){
i=1; while(i == 1){i=atomicMax(&dot_product_wr_lock,1);} //waiting for a lock to unlock, then lock
*dprod+=dprs_r4[0];
__threadfence();
i=atomicExch(&dot_product_wr_lock,0); //unlock
}
__syncthreads();
return;
}