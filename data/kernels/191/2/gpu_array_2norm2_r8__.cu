#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_2norm2_r8__(size_t arr_size, const double *arr, double *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
# arr_size - size of the array;
# arr(0:arr_size-1) - array;
OUTPUT:
# bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
size_t i,n;
double _thread_norm2;
extern __shared__ double thread_norms2_r8[];

n=gridDim.x*blockDim.x; _thread_norm2=0.0;
for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
thread_norms2_r8[threadIdx.x]=_thread_norm2;
__syncthreads();
if(threadIdx.x == 0){
bnorm2[blockIdx.x]=thread_norms2_r8[0]; for(i=1;i<blockDim.x;i++){bnorm2[blockIdx.x]+=thread_norms2_r8[i];}
}
__syncthreads();
return;
}