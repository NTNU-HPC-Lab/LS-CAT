#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_init_r4__(size_t tsize, float *arr, float val)
/** arr(:)=val **/
{
size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
size_t _gd = gridDim.x*blockDim.x;
for(size_t l=_ti;l<tsize;l+=_gd){arr[l]=val;}
return;
}