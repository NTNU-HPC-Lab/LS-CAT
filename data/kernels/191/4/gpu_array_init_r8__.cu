#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_init_r8__(size_t tsize, double *arr, double val)
/** arr(:)=val **/
{
size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
size_t _gd = gridDim.x*blockDim.x;
for(size_t l=_ti;l<tsize;l+=_gd){arr[l]=val;}
return;
}