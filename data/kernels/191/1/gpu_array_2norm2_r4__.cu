#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_2norm2_r4__(size_t arr_size, const float *arr, float *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
# arr_size - size of the array;
# arr(0:arr_size-1) - array;
OUTPUT:
# bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
size_t i,n;
float _thread_norm2;
extern __shared__ float thread_norms2_r4[];

n=gridDim.x*blockDim.x; _thread_norm2=0.0f;
for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
thread_norms2_r4[threadIdx.x]=_thread_norm2;
__syncthreads();
if(threadIdx.x == 0){
bnorm2[blockIdx.x]=thread_norms2_r4[0]; for(i=1;i<blockDim.x;i++){bnorm2[blockIdx.x]+=thread_norms2_r4[i];}
}
__syncthreads();
return;
}