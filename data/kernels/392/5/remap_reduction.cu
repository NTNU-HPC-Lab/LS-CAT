#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright Douglas Goddard 2016
// Licensed under the MIT license


// shout out to salix alba, you're a wizard mate
// http://stackoverflow.com/a/39862297/1176872
__global__ void remap_reduction( uint32_t *d_reduction, uint32_t *d_mapping, uint32_t *old_d_ij_buf, uint32_t sum_prev_size, uint32_t prev_size, uint32_t *new_d_ij_buf, uint32_t new_size)
{
uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
if(t_index < prev_size) {
if(d_reduction[t_index]) {
uint32_t index = d_mapping[t_index];
uint32_t i = *(old_d_ij_buf+2*sum_prev_size+t_index);
uint32_t j = *(old_d_ij_buf+2*sum_prev_size+prev_size+t_index);

// sort pairs in first round
if(!sum_prev_size && j < i) {
i ^= j;
j ^= i;
i ^= j;
}

*(new_d_ij_buf+2*sum_prev_size+index) = i;
*(new_d_ij_buf+2*sum_prev_size+new_size+index) = j;
}
}
}