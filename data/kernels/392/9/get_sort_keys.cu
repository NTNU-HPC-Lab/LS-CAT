#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_sort_keys( unsigned char* d_hashes, uint32_t *d_sort_keys, uint32_t *d_sort_indices, uint32_t r, uint32_t num_keys)
{
uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;
if(index < num_keys) {
d_sort_indices[index] = index;

unsigned char* input = d_hashes+index*30*sizeof(unsigned char)+3*r;
uint32_t sort_key = input[0] << 16 | input[1] << 8 | input[2];
d_sort_keys[index] = sort_key;
}
}