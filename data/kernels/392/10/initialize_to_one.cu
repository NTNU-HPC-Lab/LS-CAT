#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize_to_one( uint32_t *reduction, uint32_t size)
{
uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
if(t_index < size) {
reduction[t_index] = 1;
}
}