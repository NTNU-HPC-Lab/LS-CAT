#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expand_array( unsigned char *d_in, unsigned char *d_out)
{
uint32_t offset = blockDim.x * blockIdx.x + threadIdx.x;
unsigned char *input = d_in+offset*5*sizeof(unsigned char);
unsigned char *output = d_out+offset*6*sizeof(unsigned char);

output[0] = input[0] >> 4;
output[1] = input[0] << 4 | input[1] >> 4;
output[2] = input[1] << 4 | input[2] >> 4;
output[3] = input[2] & 0xf;
output[4] = input[3];
output[5] = input[4];
}