#include "hip/hip_runtime.h"
#include "includes.h"


static const int NTHREADS = 32;





__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel( float *gradInput, float *target, float *weights, float *total_weight, int size_average, int nframe, int ndim, int n_classes)
{
if (*total_weight <= 0) {
return;
}
int i, t;
float norm = size_average ? (1.0f / *total_weight) : 1.0f;

for (i = threadIdx.x; i < nframe; i += NTHREADS) {
t = (int)target[i] - 1;
if (t >= 0 && t < n_classes) {
gradInput[i * ndim + t] = -(weights ? weights[t] : 1.0f) * norm;
}
}
}