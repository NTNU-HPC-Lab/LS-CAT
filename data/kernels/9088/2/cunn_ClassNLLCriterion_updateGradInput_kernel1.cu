#include "hip/hip_runtime.h"
#include "includes.h"


static const int NTHREADS = 32;





__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel1( float* gradInput, float* weights, float* target, float* total_weight, int size_average, int n_classes)
{
if (*total_weight <= 0) {
return;
}
float norm = size_average ? (1.0f / *total_weight) : 1.0f;
int t = (int)*target - 1;
if (t >= 0 && t < n_classes) {
gradInput[t] = -(weights ? weights[t] : 1.0f) * norm;
}
}