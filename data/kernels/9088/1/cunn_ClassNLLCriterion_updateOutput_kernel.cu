#include "hip/hip_runtime.h"
#include "includes.h"


static const int NTHREADS = 32;





__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *total_weight, float *input, float *target, float *weights, int size_average, int nframe, int ndim, int n_classes) {
__shared__ float shInputs[NTHREADS], acc_weight[NTHREADS];
int i, t;
float cur_weight;

shInputs[threadIdx.x] = 0.0f;
acc_weight[threadIdx.x] = 0.0f;
for (i = threadIdx.x; i < nframe; i += NTHREADS) {
t = target[i] - 1;
if(t >= 0 && t < n_classes) {
cur_weight = weights ? weights[t] : 1.0f;
shInputs[threadIdx.x] -= input[i * ndim + t] * cur_weight;
acc_weight[threadIdx.x] += cur_weight;
}
}
__syncthreads();

// TODO: T4951791 Reuse code between updateOutput_kernel1 and
// updateOutput_kernel

if (threadIdx.x == 0) {
*output = *total_weight = 0;
for (i = 0; i < NTHREADS; ++i){
*output += shInputs[i];
*total_weight += acc_weight[i];
}
if (size_average && *total_weight > 0) {
*output /= *total_weight;
}
}
}