#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void average_snips(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU){

//Nfilt blocks
//Thread grid = (NrankPC, NchanNear)
//This implementation does not work correctly for real data!
//Since this_chan is function of the spike -- spikes assigned to a given template
//will have max channels that span a 2-3 channel range -- different (tidx, tidy)
//pairs can wind up trying to add to the same element of dWU, resulting in
//collisions and incorrect results. Use the single-threaded version
//average_snips_v2 instead. Speed hit is only ~ 5-6 seconds out of 360 sec for a
//typical 2 hour Neuropixels 1.0 dataset.
int my_chan, this_chan, tidx, tidy, bid, ind, Nspikes, NrankPC, NchanNear, Nchan;
float xsum = 0.0f;

Nspikes               = (int) Params[0];
NrankPC             = (int) Params[1];
Nchan                = (int) Params[7];
NchanNear             = (int) Params[6];

tidx 		= threadIdx.x;
tidy 		= threadIdx.y;
bid 		= blockIdx.x;

for(ind=0; ind<Nspikes;ind++) {
if (id[ind]==bid){
my_chan = call[ind];
this_chan = iC[tidy + NchanNear * my_chan];
xsum = uproj[tidx + NrankPC*tidy +  NrankPC*NchanNear * ind];
WU[tidx + NrankPC*this_chan + NrankPC*Nchan * bid] +=  xsum;
}
}

}