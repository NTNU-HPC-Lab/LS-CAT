#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_spikes(const double *Params, const int *id, int *nsp){

int tid, tind, bid, ind, Nspikes, Nfilters, Nthreads, Nblocks;

Nspikes               = (int) Params[0];
Nfilters             = (int) Params[2];

tid 		= threadIdx.x;
bid 		= blockIdx.x;
Nthreads              = blockDim.x;
Nblocks               = gridDim.x;

tind = tid + Nthreads *bid;

while (tind<Nfilters){
for(ind=0; ind<Nspikes;ind++)
if (id[ind]==tind)
nsp[tind] += 1;
tind += Nthreads * Nblocks;
}
}