#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bestFilter(const double *Params, const bool *iMatch, const int *Wh, const float *cmax, const float *mus, int *id, float *x){

int tid,tind,bid, my_chan, ind, Nspikes, Nfilters, Nthreads, Nchan, Nblocks;
float max_running = 0.0f;

Nspikes               = (int) Params[0];
Nfilters              = (int) Params[2];
Nchan                 = (int) Params[7];
Nthreads              = blockDim.x;
Nblocks               = gridDim.x;

tid 		= threadIdx.x;
bid 		= blockIdx.x;

tind = tid + bid * Nthreads;

while (tind<Nspikes){
max_running = mus[tind] * mus[tind];
id[tind] = 0;
my_chan = Wh[tind];
for(ind=0; ind<Nfilters; ind++)
if (iMatch[my_chan + ind * Nchan])
if (cmax[tind + ind*Nspikes] < max_running){
id[tind] = ind;
max_running = cmax[tind + ind*Nspikes];
}
x[tind] = max_running;
tind += Nblocks*Nthreads;
}

}