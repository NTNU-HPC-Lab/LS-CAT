#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bestFilter(const double *Params,  const bool *match, const int *iC, const int *call, const float *cmax, int *id, float *cx){

int Nchan, tid,tind,bid, ind, Nspikes, Nfilters, Nthreads, Nblocks, my_chan;
float max_running = 0.0f;

Nspikes               = (int) Params[0];
Nfilters              = (int) Params[2];
Nthreads              = blockDim.x;
Nblocks               = gridDim.x;
Nchan                = (int) Params[7];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

tind = tid + bid * Nthreads;

while (tind<Nspikes){
max_running = 0.0f;
id[tind] = 0;
my_chan = call[tind];

for(ind=0; ind<Nfilters; ind++)
if (match[my_chan + ind * Nchan])
if (cmax[tind + ind*Nspikes] > max_running){
id[tind] = ind;
max_running = cmax[tind + ind*Nspikes];
}


cx[tind] = max_running;

tind += Nblocks*Nthreads;
}
}