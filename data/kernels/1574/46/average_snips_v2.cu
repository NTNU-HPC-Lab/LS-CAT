#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void average_snips_v2(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU){


// jic, version with no threading over features, to avoid
// collisions when summing WU
// run

int my_chan, this_chan, bid, ind, Nspikes, NrankPC, NchanNear, Nchan;
float xsum = 0.0f;
int chanIndex, pcIndex;

Nspikes               = (int) Params[0];
NrankPC             = (int) Params[1];
Nchan                = (int) Params[7];
NchanNear             = (int) Params[6];


bid 		= blockIdx.x;

for(ind=0; ind<Nspikes;ind++)
if (id[ind]==bid){
my_chan = call[ind];
for (chanIndex = 0; chanIndex < NchanNear; ++chanIndex) {
this_chan = iC[chanIndex + NchanNear * my_chan];
for (pcIndex = 0; pcIndex < NrankPC; ++pcIndex) {
xsum = uproj[pcIndex + NrankPC*chanIndex +  NrankPC*NchanNear * ind];
WU[pcIndex + NrankPC*this_chan + NrankPC*Nchan * bid] +=  xsum;
}
}

}
}