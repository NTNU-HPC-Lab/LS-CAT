#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_spikes(const double *Params, const int *id, int *nsp, const float *x, float *V){

int tid, tind, bid, ind, Nspikes, Nfilters, NthreadsMe, Nblocks;

Nspikes               = (int) Params[0];
Nfilters             = (int) Params[2];

tid 		= threadIdx.x;
bid 		= blockIdx.x;
NthreadsMe              = blockDim.x;
Nblocks               = gridDim.x;

tind = tid + NthreadsMe *bid;

while (tind<Nfilters){
for(ind=0; ind<Nspikes;ind++)
if (id[ind]==tind){
nsp[tind] ++;
V[tind] += x[tind];
}
V[tind] = V[tind] / (.001f + (float) nsp[tind]);

tind += NthreadsMe * Nblocks;
}


}