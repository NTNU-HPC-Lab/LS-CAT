#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void average_snips(const double *Params, const int *ioff, const int *id, const float *uproj, const float *cmax, const int *iList, float *cf, float *WU){

int tid, bid, ind, Nspikes, Nfeatures, NfeatW, Nnearest, t;
float xsum = 0.0f, pm;

Nspikes               = (int) Params[0];
Nfeatures             = (int) Params[1];
pm                    = (float) Params[3];
NfeatW                = (int) Params[4];
Nnearest              = (int) Params[6];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

for(ind=0; ind<Nspikes;ind++)
if (id[ind]==bid){

xsum = uproj[tid + Nfeatures * ind];
WU[tid + ioff[ind] + NfeatW * bid] = pm * WU[tid + ioff[ind] + NfeatW * bid]
+ (1-pm) * xsum;

// go through the top 10 nearest filters and match them
for (t=0;t<Nnearest;t++)
cf[ind + t*Nspikes] = cmax[ind + Nspikes * iList[t + Nnearest*bid]];

}
}