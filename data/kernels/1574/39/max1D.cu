#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void	max1D(const double *Params, const float *data, float *conv_sig){

volatile __shared__ float  sdata[Nthreads+81];
float y, spkTh;
int tid, tid0, bid, i, NT, nt0;

NT 		= (int) Params[0];
nt0       = (int) Params[3];
spkTh    = (float) Params[5];
tid 		= threadIdx.x;
bid 		= blockIdx.x;

tid0 = 0;
while (tid0<NT-Nthreads-nt0+1){
if (tid<nt0)
sdata[tid]   = data[tid0 + tid + NT*bid];
sdata[tid + nt0] = data[nt0+tid0 + tid+ NT*bid];
__syncthreads();

y = 0.0f;
#pragma unroll 4
for(i=0;i<nt0;i++)
y    = max(y, sdata[tid+i]);

if (y>spkTh)
conv_sig[tid0  + tid + NT*bid]   = y;

tid0+=Nthreads;
__syncthreads();
}
}