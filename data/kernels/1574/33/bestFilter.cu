#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bestFilter(const double *Params, const bool *iW, const float *cmax, int *id){

int tid,tind,bid, ind, Nspikes, Nfilters, Nthreads, Nblocks;
float max_running = 0.0f, Th;

Nspikes               = (int) Params[0];
Nfilters              = (int) Params[2];
Nthreads              = blockDim.x;
Nblocks               = gridDim.x;
Th                    = (float) Params[7];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

tind = tid + bid * Nthreads;

while (tind<Nspikes){
max_running = 0.0f;
id[tind] = 0;

for(ind=0; ind<Nfilters; ind++)
if (iW[tind + ind*Nspikes])
if (cmax[tind + ind*Nspikes] > max_running){
id[tind] = ind;
max_running = cmax[tind + ind*Nspikes];
}

if (max_running < Th*Th)
id[tind] = -1;

tind += Nblocks*Nthreads;
}
}