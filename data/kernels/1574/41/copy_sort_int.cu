#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_sort_int( const int *orig, const unsigned int *sort_idx, const unsigned int nitems, int *sorted ) {
for( int i = 0; i < nitems; ++ i ) {
sorted[sort_idx[i]] = orig[i];
}
}