#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_dWU(const double *Params, const float *bigArray, float *WU) {

int tid,bid, ind, Nfilters, Nthreads, Nfeatures, Nblocks, NfeatW, nWU, nElem;
float sum = 0.0f;

Nfeatures             = (int) Params[1];  //NrankPC, number of pcs
NfeatW                = (int) Params[4];  //Nchan*nPC
Nfilters              = (int) Params[2];
Nthreads              = blockDim.x;
Nblocks               = gridDim.x;

tid 		= threadIdx.x;
bid 		= blockIdx.x;


//WU is NfeatW x Nfilters.

nWU = NfeatW * Nfilters;
nElem = Nfeatures*NfeatW; //number of elements in each subArray of bigArray

//Calculate which element we're addressing
int tind = tid + bid * Nthreads;

int currFilt, currFW, currIndex;
while (tind < nWU){


//which filter and element of WU?
currFilt = floor((double)(tind/NfeatW));
currFW = tind - currFilt*NfeatW;

//Sum up the Nfeature elements of bigArray that correspond to this
//filter and NfeatW

sum = 0.0f;

for(ind=0; ind<Nfeatures; ind++) {
//bigArray is Nfilter arrays of Nfeature x NfeatW;
currIndex = currFilt*nElem + ind*NfeatW + currFW;
sum += bigArray[ currIndex ];
}

WU[tind] += sum;
tind += Nblocks*Nthreads;

}

}