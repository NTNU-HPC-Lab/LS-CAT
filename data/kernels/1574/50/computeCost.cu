#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCost(const double *Params, const float *Ws, const float *mus, const float *W, const float *mu, const bool *iMatch, const int *iC, const int *Wh, float *cmax){

int j, tid, bid, Nspikes, my_chan, this_chan, Nchan, NrankPC, NchanNear, Nthreads, k;
float xsum = 0.0f, Ci;

Nspikes               = (int) Params[0];  //more accurately, number of comparisons, Nfilt*Nbatch
Nchan                 = (int) Params[7];
NrankPC                 = (int) Params[1];
NchanNear                 = (int) Params[6];
Nthreads              = blockDim.x;


tid 		= threadIdx.x;
bid 		= blockIdx.x;

while(tid<Nspikes){
my_chan = Wh[tid];
if (iMatch[my_chan + bid*Nchan]){
xsum = 0.0f;
for (k=0;k<NchanNear;k++){
this_chan = iC[k + NchanNear * my_chan];
for (j=0;j<NrankPC;j++)
xsum += Ws[j + NrankPC*k + NrankPC*NchanNear * tid] *
W[j + NrankPC*this_chan + NrankPC*Nchan * bid];

}

Ci = mu[bid]*mu[bid] + mus[tid]*mus[tid] -2*mus[tid]*mu[bid]*xsum;
cmax[tid + bid*Nspikes] = Ci;
}
tid+= Nthreads;
}
}