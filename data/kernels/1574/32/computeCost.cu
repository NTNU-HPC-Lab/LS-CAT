#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, const int *ioff, const bool *iW, float *cmax){

int tid, bid, Nspikes, Nfeatures, NfeatW, Nthreads, k;
float xsum = 0.0f, Ci, lam;

Nspikes               = (int) Params[0];
Nfeatures             = (int) Params[1];
NfeatW                = (int) Params[4];
Nthreads              = blockDim.x;
lam                   = (float) Params[5];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

while(tid<Nspikes){
if (iW[tid + bid*Nspikes]){
xsum = 0.0f;
for (k=0;k<Nfeatures;k++)
xsum += uproj[k + Nfeatures * tid] * W[k + ioff[tid] +  NfeatW * bid];

Ci = max(0.0f, xsum) + lam/mu[bid];

cmax[tid + bid*Nspikes] = Ci * Ci / (1.0f + lam/(mu[bid] * mu[bid])) - lam;
}
tid+= Nthreads;
}

}