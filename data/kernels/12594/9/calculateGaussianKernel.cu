#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateGaussianKernel(float *gaussKernel, const float sigma, int halfKernelWidth){

/// pixel index of this thread
/// this makes the normal curve
int i = threadIdx.x - halfKernelWidth;
extern __shared__ float s_gaussKernel[];
__shared__ float sum;

/// this kernel must allocate 'kernelWidth' threads
s_gaussKernel[threadIdx.x] = (__fdividef(1,(sqrtf(2*M_PI*sigma))))*expf((-1)*(__fdividef((i*i),(2*sigma*sigma))));

__syncthreads();

/// Thread 0 sum all the gassian kernel array
// This is not so bad because the array is always short
if (!threadIdx.x) {
int th;
sum = 0;
for(th = 0; th<blockDim.x; th++) sum += s_gaussKernel[th];
}

__syncthreads();

gaussKernel[threadIdx.x] = s_gaussKernel[threadIdx.x]/sum;

}