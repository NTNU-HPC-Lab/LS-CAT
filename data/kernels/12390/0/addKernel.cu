#include "hip/hip_runtime.h"
#include "includes.h"




//#define array_size 100000000
#define array_size 101

//987459712


hipError_t addWithCuda(int *total);

__shared__ int temp[array_size];

__global__ void addKernel(int *tid_c, int *tid_total)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
tid_c[tid] = tid;
if (tid <= array_size)
{

temp[threadIdx.x] = tid;

if (threadIdx.x==0)
{
for(int i=0;i<=blockDim.x;i++)
{
//__syncthreads();
atomicAdd(tid_total, temp[i]);
//__syncthreads();
//printf("i = %d \n", *tid_total);
}
}

}

}