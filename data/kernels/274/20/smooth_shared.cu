#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_shared(float * v_new, const float * v) {
extern __shared__ float s[];
int id = blockDim.x * blockIdx.x + threadIdx.x;
s[threadIdx.x + 1] = v[id];

if (threadIdx.x == 0) {
int start = blockDim.x * blockIdx.x;
int left = max(0, start - 1);
s[0] = v[left];
int end = blockDim.x * gridDim.x;
int right = min(end - 1, blockDim.x * blockIdx.x + blockDim.x);
s[blockDim.x + 1] = v[right];
}

__syncthreads();

int tid = threadIdx.x + 1;
v_new[id] = 0.25f * s[tid - 1] + 0.5f * s[tid] + 0.25f * s[tid + 1];
}