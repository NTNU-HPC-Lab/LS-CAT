#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_normalized_cross_correlation( float*         d_response, unsigned char* d_original, unsigned char* d_template, int            num_pixels_y, int            num_pixels_x, int            template_half_height, int            template_height, int            template_half_width, int            template_width, int            template_size, float          template_mean )
{
int  ny             = num_pixels_y;
int  nx             = num_pixels_x;
int  knx            = template_width;
int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

if ( image_index_2d.x < nx && image_index_2d.y < ny )
{
//
// compute image mean
//
float image_sum = 0.0f;

for ( int y = -template_half_height; y <= template_half_height; y++ )
{
for ( int x = -template_half_width; x <= template_half_width; x++ )
{
int2 image_offset_index_2d         = make_int2( image_index_2d.x + x, image_index_2d.y + y );
int2 image_offset_index_2d_clamped = make_int2( min( nx - 1, max( 0, image_offset_index_2d.x ) ), min( ny - 1, max( 0, image_offset_index_2d.y ) ) );
int  image_offset_index_1d_clamped = ( nx * image_offset_index_2d_clamped.y ) + image_offset_index_2d_clamped.x;

unsigned char image_offset_value = d_original[ image_offset_index_1d_clamped ];

image_sum += (float)image_offset_value;
}
}

float image_mean = image_sum / (float)template_size;

//
// compute sums
//
float sum_of_image_template_diff_products = 0.0f;
float sum_of_squared_image_diffs          = 0.0f;
float sum_of_squared_template_diffs       = 0.0f;

for ( int y = -template_half_height; y <= template_half_height; y++ )
{
for ( int x = -template_half_width; x <= template_half_width; x++ )
{
int2 image_offset_index_2d         = make_int2( image_index_2d.x + x, image_index_2d.y + y );
int2 image_offset_index_2d_clamped = make_int2( min( nx - 1, max( 0, image_offset_index_2d.x ) ), min( ny - 1, max( 0, image_offset_index_2d.y ) ) );
int  image_offset_index_1d_clamped = ( nx * image_offset_index_2d_clamped.y ) + image_offset_index_2d_clamped.x;

unsigned char image_offset_value = d_original[ image_offset_index_1d_clamped ];
float         image_diff         = (float)image_offset_value - image_mean;

int2 template_index_2d = make_int2( x + template_half_width, y + template_half_height );
int  template_index_1d = ( knx * template_index_2d.y ) + template_index_2d.x;

unsigned char template_value = d_template[ template_index_1d ];
float         template_diff  = template_value - template_mean;

float image_template_diff_product = image_offset_value   * template_diff;
float squared_image_diff          = image_diff           * image_diff;
float squared_template_diff       = template_diff        * template_diff;

sum_of_image_template_diff_products += image_template_diff_product;
sum_of_squared_image_diffs          += squared_image_diff;
sum_of_squared_template_diffs       += squared_template_diff;
}
}


//
// compute final result
//
float result_value = 0.0f;

if ( sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0 )
{
result_value = sum_of_image_template_diff_products / sqrt( sum_of_squared_image_diffs * sum_of_squared_template_diffs );
}

d_response[ image_index_1d ] = result_value;
}
}