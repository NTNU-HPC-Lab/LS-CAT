#include "hip/hip_runtime.h"
#include "includes.h"

// Subpart A:
// Write step 1 as a kernel that operates on threads 0--31.
// Assume that the input flags are 0 for false and 1 for true and are stored
// in a local per-thread register called p (for predicate).
//
// You have access to 31 words of shared memory s[0:31], with s[0]
// corresponding to thread 0 and s[31] corresponding to thread 31.
// You may change the values of s[0:31]. Put the return sum in s[0].
// Your code should execute no more than 5 warp-wide addition operations.

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
// Assumes values in 'p' are either 1 or 0
// Assumes s[0:31] are allocated
// Sums p across warp, returning the result. Suggest you put
// result in s[0] and return it
// You may change any value in s
// You should execute no more than 5 + operations (if you're doing
// 31, you're doing it wrong)

int tid = threadIdx.x;
s[tid] = p;
__syncthreads();
for (int i = blockDim.x / 2; i > 0; i >>= 1) { // This could be unrolled
if (tid < i) {
s[tid] += s[tid+i];
}
__syncthreads();
}
return s[0];
}
__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
extern __shared__ unsigned int s[];
int t = threadIdx.x;
int p = d_in[t];
unsigned int sr = shared_reduce(p, s);
if (t == 0)
{
*d_out_shared = sr;
}
}