#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb_to_xyY( float* d_r, float* d_g, float* d_b, float* d_x, float* d_y, float* d_log_Y, float  delta, int    num_pixels_y, int    num_pixels_x )
{
int  ny             = num_pixels_y;
int  nx             = num_pixels_x;
int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

if ( image_index_2d.x < nx && image_index_2d.y < ny )
{
float r = d_r[ image_index_1d ];
float g = d_g[ image_index_1d ];
float b = d_b[ image_index_1d ];

float X = ( r * 0.4124f ) + ( g * 0.3576f ) + ( b * 0.1805f );
float Y = ( r * 0.2126f ) + ( g * 0.7152f ) + ( b * 0.0722f );
float Z = ( r * 0.0193f ) + ( g * 0.1192f ) + ( b * 0.9505f );

float L = X + Y + Z;
float x = X / L;
float y = Y / L;

float log_Y = log10f( delta + Y );

d_x[ image_index_1d ]     = x;
d_y[ image_index_1d ]     = y;
d_log_Y[ image_index_1d ] = log_Y;
}
}