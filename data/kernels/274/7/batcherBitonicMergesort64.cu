#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void swap(float& a, float& b)
{
float temp = a;
a = b;
b = temp;
}
__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
// you are guaranteed this is called with <<<1, 64, 64*4>>>
extern __shared__ float sdata[];
int tid  = threadIdx.x;
sdata[tid] = d_in[tid];
__syncthreads();

for (int stage = 0; stage <= 5; stage++)
{
for (int substage = stage; substage >= 0; substage--)
{
int distance = 1 << substage; // Distance to value to be compared
int comparison = tid - distance; // Value to be compared
int div = 1 << (stage + 1);
// Skip values that should not be compared
if (comparison < 0 || (comparison / div) != (tid / div)) {
continue;
}
bool up = (comparison / div) % 2 == 1;
if (up) {
if (sdata[tid] > sdata[comparison]) {
swap(sdata[tid], sdata[comparison]);
}
} else {
if (sdata[tid] < sdata[comparison]) {
swap(sdata[tid], sdata[comparison]);
}
}
}
__syncthreads();
}

d_out[tid] = sdata[tid];
}