#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < num_rows;
i += blockDim.y * gridDim.y) {
for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < num_cols;
j += blockDim.x * gridDim.x) {
float f = feats[i * ldf + j];
feats_sq[i * lds + j] = f * f;
}
}
}