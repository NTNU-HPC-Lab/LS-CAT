#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
for (int32_t i = threadIdx.x; i < col_src; i += blockDim.x) {
*(dst + blockIdx.x * col_dst + i) = *(src + blockIdx.x * col_src + i);
}
}