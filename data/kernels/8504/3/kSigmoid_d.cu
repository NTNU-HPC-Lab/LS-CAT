#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSigmoid_d(const int nThreads, float const *input, float *output) {

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = input[i] * (1 - input[i]);
}
}