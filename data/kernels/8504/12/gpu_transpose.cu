#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpose(const float* src, float* dst, int colssrc, int colsdst, int n) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while (tid < n) {
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = cdst;
int csrc = rdst;
dst[tid] = src[rsrc * colssrc + csrc];
tid += stride;
}
}