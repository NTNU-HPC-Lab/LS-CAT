#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_multiply(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

float CValue = 0;

int Row = blockIdx.y*TILE_DIM + threadIdx.y;
int Col = blockIdx.x*TILE_DIM + threadIdx.x;

__shared__ float As[TILE_DIM][TILE_DIM];
__shared__ float Bs[TILE_DIM][TILE_DIM];

for (int k = 0; k < (TILE_DIM + ACols - 1) / TILE_DIM; k++) {

if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
As[threadIdx.y][threadIdx.x] = A[Row*ACols + k * TILE_DIM + threadIdx.x];
else
As[threadIdx.y][threadIdx.x] = 0.0;

if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
else
Bs[threadIdx.y][threadIdx.x] = 0.0;
__syncthreads();

for (int n = 0; n < TILE_DIM; ++n)
CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

__syncthreads();
}

if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue;
}