#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTanhDerivative(const int nThreads, float const *input, float *output)
{

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = 1 - (tanh(input[i]) * tanh(input[i]));
}
}