#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSigmoid(const int nThreads, float const *input, float *output) {

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = 1.0 / (1.0 + std::exp(-input[i]));
}
}