#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_add(float* first, float* second, size_t sizeFirst)
{
int threadId = threadIdx.x + blockIdx.x * blockDim.x;

while (threadId < sizeFirst) {
first[threadId] = (first[threadId] +  second[threadId]);
threadId += ( blockDim.x * gridDim.x );
}
}