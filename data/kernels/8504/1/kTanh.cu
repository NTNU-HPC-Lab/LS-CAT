#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTanh(const int nThreads, float const *input, float *output) {

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = tanh(input[i]);
}
}