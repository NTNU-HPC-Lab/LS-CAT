#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mull2(float* a, float* b, float* c, int n, int m,int p)
{
int i = blockIdx.x * 32 + threadIdx.x;
int j = blockIdx.y;

float sum = 0.0f;
for (int k = 0; k < p; ++k) {
sum += b[i + n * k] * c[k + p * j];

}
a[i + n * j] = sum;
}