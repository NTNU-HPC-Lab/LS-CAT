#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows, const int m1_columns, const int m2_columns) {

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
int r = (int)i / m2_columns;
int c = i % m2_columns;
float t_output = 0.f;

for (int k = 0; k < m1_columns; ++k) {
t_output += m1[r * m1_columns + k] * m2[k * m2_columns + c];
}

output[i] = t_output;
}
}