#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
int g_tId = threadIdx.x + blockDim.x * blockIdx.x;
unsigned int warpId = threadIdx.x / warpSize;

if ( g_tId < vectorSize) {
c[g_tId] = a[g_tId] + b[g_tId];
printf("thread id %d , warp Id %d , block id %d\n", g_tId, warpId,blockIdx.x);
}
}