#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA4(float *C, float *A, float *B, unsigned int n)
{
/*
Each block computes a tile
*/
int tileWidth = 32;

// Define the starting row and ending row for each thread
int startRow = blockIdx.y * blockDim.y + threadIdx.y * tileWidth;
int endRow = startRow + tileWidth;

// Define the starting column and ending column for each thread
int startCol = blockIdx.x * blockDim.x + threadIdx.x * tileWidth;
int endCol = startCol + tileWidth;

// Now we have some block in 2 dimensions
for (int row = startRow; row < endRow; row++) {
for (int col = startCol; col < endCol; col++) {

if (row >= n || col >= n) {
continue;
}

// Compute the proper sum for each block
float sum = 0.0f;	// Defined as a register (Better than directly writing to C)
for (int k = 0; k < n; k++) {
sum += A[row * n + k] * B[k * n + col];
}

// Write back sum into C
C[row * n + col] = sum;
}
}
}