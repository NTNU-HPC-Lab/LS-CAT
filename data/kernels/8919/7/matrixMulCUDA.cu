#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA(float *C, float *A, float *B, int n)
{
int k;

// Get the row and the column in which thread resides in a block
int row = threadIdx.x;
int col = threadIdx.y;
float sum = 0.0f;
if (row >= n || col >= n) {
return;
}
for (k = 0; k < n; k++) {
sum += A[row * n + k] * B[k * n + col];

}
C[row * n + col] = sum;
}