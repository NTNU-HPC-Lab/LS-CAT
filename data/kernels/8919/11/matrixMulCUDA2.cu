#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA2(float *C, float *A, float *B, int n)
{
/*
Each thread computes more than 1 matrix elements
*/

// Define the starting row and ending row for each thread
int startRow = threadIdx.y * TILE_WIDTH;
int endRow = startRow + TILE_WIDTH;

// Define the starting column and ending column for each thread
int startCol = threadIdx.x * TILE_WIDTH;
int endCol = startCol + TILE_WIDTH;

// Now we have some block in 2 dimensions
for (int row = startRow; row < endRow; row++) {
for (int col = startCol; col < endCol; col++) {

// Compute the proper sum for each block
float sum = 0.0f;	// Defined as a register (Better than directly writing to C)
for (int k = 0; k < n; k++) {
sum += A[row * n + k] * B[k * n + col];
}

// Write back sum into C
C[row * n + col] = sum;
}
}
}