#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA5(float *C, float *A, float *B, unsigned int n)
{

const int tileWidth = 1;

// Define the starting row and ending row for each thread block
int startRow = blockIdx.y * blockDim.y + threadIdx.y * tileWidth;
int endRow = startRow + tileWidth;

// Define the starting column and ending column for each thread block
int startCol = blockIdx.x * blockDim.x + threadIdx.x * tileWidth;
int endCol = startCol + tileWidth;

// Each block of threads allocate space on shared memory
__shared__ float A_S[32 * 32 * 4];
__shared__ float B_S[32 * 32 * 4];

// Each thread helps copying the proper indexes into the shared memory
// Now we have some blocks in 2 dimensions
for (int row = startRow; row < endRow; row++) {
for (int col = startCol; col < endCol; col++) {



// Copy data into shared memory
for (int k = 0; k < n; k++) {
A_S[k] = A[row * n + k];
B_S[k] = B[k * n + col];
}

// Synchronize all threads to make a tile completely ready to go!
__syncthreads();

// Compute the proper sum for each block
float sum = 0.0f;	// Defined as a register (Better than directly writing to C)
for (int k = 0; k < n; k++) {
sum += A_S[k] * B_S[k];
}

// Write back sum into C
C[row * n + col] = sum;
}
}
}