#include "hip/hip_runtime.h"
#include "includes.h"

// GPU Libraries

// Macro to handle errors occured in CUDA api
__device__ void recursiveReduce(int *g_inData, int *g_outData, int inSize, int outSize)
{
extern __shared__ int sData[];

// Identification
unsigned int tId = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

// Initialize
sData[tId] = 0;

__syncthreads();

// Fill up the shared memory
if (tId < blockDim.x) {
sData[tId] = g_inData[i];
}

__syncthreads();

// Tree based reduction
for (unsigned int d = 1; d < blockDim.x; d *= 2) {
if (tId % (2 * d) == 0)
if (tId + d < blockDim.x)
sData[tId] += sData[tId + d];

__syncthreads();
}

// Write the result for this block to global memory
if (tId == 0) {
g_outData[blockIdx.x] = sData[0];
}


__syncthreads();

// Recursive call
if (outSize > 1 && i == 0) {

// Kernel Launch
recursiveReduce(g_outData, g_outData, outSize, (outSize - 1) / blockDim.x + 1);

}
else return;

}
__global__ void reduceKernel(int *g_inData, int *g_outData, int inSize, int outSize)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i == 0) {
recursiveReduce(g_inData, g_outData, inSize, outSize);
}
}