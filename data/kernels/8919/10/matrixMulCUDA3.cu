#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA3(float *C, float  *B, float *A, int n)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x+ threadIdx.x;

float sum = 0.0f;

if (row >= n || col >= n) {
return;
}

for (int k = 0; k < n; k++) {
sum += A[row * n + k] * B[k * n + col];
}
C[row * n + col] = sum;
}