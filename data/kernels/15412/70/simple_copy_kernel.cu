#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size)
dst[index] = src[index];
}