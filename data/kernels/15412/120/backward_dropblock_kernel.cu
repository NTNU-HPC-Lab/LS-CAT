#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_dropblock_kernel(float *pass, float *delta, int size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= size) return;

if (pass[index] == 0) delta[index] = 0;
}