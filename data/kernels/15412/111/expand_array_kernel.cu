#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;

if (index < current_size) {
for (int i = 0; i < groups; ++i) {
dst_gpu[index + i*current_size] = src_gpu[index];
}
}
}