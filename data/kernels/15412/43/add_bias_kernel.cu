#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias_kernel(float *output, float *biases, int batch, int filters, int spatial, int current_size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= current_size) return;

int f = (index / spatial) % filters;
output[index] += biases[f];
}