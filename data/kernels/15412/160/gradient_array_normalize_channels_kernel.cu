#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *delta_gpu)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

int wh_i = i % wh_step;
int b = i / wh_step;

if (i < size) {
int k;
/*
float grad = 0;
for (k = 0; k < channels; ++k) {
const int index = wh_i + k * wh_step + b*wh_step*channels;
float out = x[index];
float delta = delta_gpu[index];
grad += out*fabs(delta);
}
*/
for (k = 0; k < channels; ++k) {
const int index = wh_i + k * wh_step + b*wh_step*channels;
if (x[index] > 0) {
float delta = delta_gpu[index];
float grad = x[index];
delta = delta * grad;
delta_gpu[index] = delta;
}
}
}
}