#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_of_mults_kernel(float *a1, float *a2, float *b1, float *b2, size_t size, float *dst)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
dst[index] = a1[index] * a2[index] + b1[index] * b2[index];
}
}