#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void reduce_and_expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;

if (index < current_size) {
float val = 0;
for (int i = 0; i < groups; ++i) {
val += src_gpu[index + i*current_size];
}
for (int i = 0; i < groups; ++i) {
dst_gpu[index + i*current_size] = val / groups;
}
}
}