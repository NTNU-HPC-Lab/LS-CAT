#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void stretch_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, float scale, int reverse)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
const int kernel_area = kernel_size * kernel_size;
const int i = index * kernel_area;

const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
const int stage_id = index / stage_step;

// nweights = (c / groups) * n * size * size;
// kernel_area = size*size

if (i < nweights)
{

if (stage_id == 0) {
// simple copy
for (int x = 0; x < kernel_size; ++x) {
for (int y = 0; y < kernel_size; ++y) {
weight_deform_gpu[x + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
}
}
}
else if (stage_id > 0)
{
if (stage_id == 1) scale = 0.65;
else if (stage_id == 2) scale = 0.8;
else if (stage_id == 3) scale = 1.3;

if (reverse) scale = 1 / scale;

const int x_c = kernel_size / 2;
const int y_c = kernel_size / 2;

float dropout_sum = 0;

for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
// Xsource = x_c + (x_d - x_c) / scale
// Ysource = y_c + (y_d - y_c) / scale

float x_s = x_c + (x - x_c) / scale;
float y_s = y_c + (y - y_c) / scale;

int x_0 = floor(x_s);   // round down
int x_1 = ceil(x_s);    // round up
if (x_0 == x_1) x_1 = x_0 + 1;
int y_0 = floor(y_s);
int y_1 = ceil(y_s);
if (y_0 == y_1) y_1 = y_0 + 1;

float c_x_0 = x_1 - x_s;
float c_x_1 = x_s - x_0;
float c_y_0 = y_1 - y_s;
float c_y_1 = y_s - y_0;

float val = 0;
if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
else dropout_sum += c_x_0 * c_y_0;

if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
else dropout_sum += c_x_1 * c_y_0;

if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
else dropout_sum += c_x_0 * c_y_1;

if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
else dropout_sum += c_x_1 * c_y_1;

weight_deform_gpu[x + y*kernel_size + i] = val;
}
}

// compensate for dropped items
//const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
//if (scale < 1) weight_deform_gpu[x + y*kernel_size + i] /= scale;// *= coef;
weight_deform_gpu[x + y*kernel_size + i] /= scale;// *= coef;
}
}
}
}
}