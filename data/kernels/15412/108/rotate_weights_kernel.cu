#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void rotate_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, int reverse)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
const int kernel_area = kernel_size * kernel_size;
const int i = index * kernel_area;

const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
const int stage_id = index / stage_step;

// nweights = (c / groups) * n * size * size;
// kernel_area = size*size

if (i < nweights)
{
// if(reverse)

if (stage_id == 0) {
// simple copy
for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
const int src_i = x + y*kernel_size + i;
const int dst_i = x + y*kernel_size + i;
if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
}
}
}
else if (stage_id == 1)
{
// 90 degree clockwise rotation - 1
for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
const int src_i = x + y*kernel_size + i;
const int dst_i = (kernel_size - 1 - y) + x*kernel_size + i;
if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
}
}
}
else if (stage_id == 2)
{
// 180 degree clockwise rotation - 2
for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
const int src_i = x + y*kernel_size + i;
const int dst_i = (kernel_size - 1 - x) + (kernel_size - 1 - y)*kernel_size + i;
if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
}
}
}
else if (stage_id == 3)
{
// 270 degree clockwise rotation - 3
for (int y = 0; y < kernel_size; ++y) {
for (int x = 0; x < kernel_size; ++x) {
const int src_i = x + y*kernel_size + i;
const int dst_i = y + (kernel_size - 1 - x)*kernel_size + i;
if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
}
}
}
}
}