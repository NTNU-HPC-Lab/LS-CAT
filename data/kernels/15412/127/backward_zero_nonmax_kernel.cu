#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_zero_nonmax_kernel(int n, int *indexes, float *prev_delta)
{

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) return;

if (indexes[id] != id) prev_delta[id] = 0;
}