#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void mult_inverse_array_kernel(const float *src_gpu, float *dst_gpu, int size, const float eps)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;

if (index < size) {
float val = src_gpu[index];
float sign = (val < 0) ? -1 : 1;
// eps = 1 by default
// eps = 2 - lower delta
// eps = 0 - higher delta (linear)
// eps = -1 - high delta (inverse number)
dst_gpu[index] = powf(fabs(val), eps) * sign;
}
}