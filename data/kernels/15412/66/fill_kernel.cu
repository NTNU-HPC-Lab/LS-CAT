#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= N) return;
X[index*INCX] = ALPHA;
}