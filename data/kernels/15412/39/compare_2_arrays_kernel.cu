#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compare_2_arrays_kernel(float *one, float *two, int size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= size) return;

const float diff = 100 * fabs(one[index] - two[index]) / fabs(one[index]);

if (diff > 10) printf(" i: %d - one = %f, two = %f, diff = %f %% \n", index, one[index], two[index], diff);
}