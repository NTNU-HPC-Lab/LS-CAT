#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_min_max_kernel(int N, float MIN, float MAX, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < N) X[i*INCX] = fminf(MAX, fmaxf(MIN, X[i*INCX]));
}