#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_dropblock_kernel(float *output, int size, int outputs, float *drop_blocks_scale)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= size) return;

const int b = index / outputs;
output[index] *= drop_blocks_scale[b];
}