#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float softplus_kernel(float x, float threshold = 20) {
if (x > threshold) return x;                // too large
else if (x < -threshold) return expf(x);    // too small
return log1pf(expf(x));
//return logf(expf(x) + 1);
}
__global__ void gradient_array_mish_kernel(int n, float *activation_input_gpu, float *delta)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
const float MISH_THRESHOLD = 20.0f;

// implementation from TensorFlow: https://github.com/tensorflow/addons/blob/093cdfa85d334cbe19a37624c33198f3140109ed/tensorflow_addons/custom_ops/activations/cc/kernels/mish_op.h#L66-L80
// implementation from Pytorch: https://github.com/thomasbrandon/mish-cuda/blob/master/csrc/mish.h#L26-L31
// log1p(x) == log(x + 1)
const float inp = activation_input_gpu[i];
const float sp = softplus_kernel(inp, MISH_THRESHOLD);
const float grad_sp = -expm1f(-sp);
//const float grad_sp = 1 - expf(-sp);
const float tsp = tanh(sp);
const float grad_tsp = (1 - tsp*tsp) * grad_sp;
const float grad = inp * grad_tsp + tsp;
delta[i] *= grad;

//float x = activation_input[i];
//float d = 2 * expf(x) + expf(2 * x) + 2;
//float w = 4 * (x + 1) + 4 * expf(2 * x) + expf(3 * x) + expf(x)*(4 * x + 6);
//float derivative = expf(x) * w / (d * d);
//delta[i] *= derivative;
}
}