#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float hard_mish_yashas_grad(float x)
{
if (x > 0)
return 1;
if (x > -2)
return x + 1;
return 0;
}
__device__ float hard_mish_yashas(float x)
{
if (x > 0)
return x;
if (x > -2)
return x * x / 2 + x;
return 0;
}
__device__ float mish_yashas(float x)
{
float e = __expf(x);
if (x <= -18.0f)
return x * e;

float n = e * e + 2 * e;
if (x <= -5.0f)
return x * __fdividef(n, n + 2);

return x - 2 * __fdividef(x, n + 2);
}
__global__ void gradient_array_hard_mish_kernel(int n, float *activation_input_gpu, float *delta)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {

const float x = activation_input_gpu[i];
delta[i] *= hard_mish_yashas_grad(x);
}
}