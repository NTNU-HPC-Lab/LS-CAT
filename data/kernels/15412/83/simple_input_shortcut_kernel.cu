#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= size) return;

out[id] = in[id] + add[id];
}