#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
return ~(a^b) & 0b1;
}
__device__ __host__ static inline unsigned char get_bit(unsigned char const*const src, size_t index) {
size_t src_i = index / 8;
int src_shift = index % 8;
unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
//unsigned char val = (src[src_i] & (1 << (8 - src_shift))) > 0;
return val;
}
__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;

int fil;
// filter index
//for (fil = 0; fil < n; ++fil)
int chan, y, x, f_y, f_x;
// channel index
//for (chan = 0; chan < in_c; ++chan)
// input - y
//for (y = 0; y < in_h; ++y)
// input - x
//for (x = 0; x < in_w; ++x)
x = index % in_w;
int index2 = index / in_w;
y = index2 % in_h;
fil = index2 / in_h;
if (fil < n)    // (1-6 for one BLOCK)
{
//float mean_val = mean_arr_gpu[fil];
int const output_index = fil*in_w*in_h + y*in_w + x;
int sum = 0;
int good_val = 0;

for (chan = 0; chan < in_c; ++chan)
{
//int const weights_pre_index = fil*in_c*size*size + chan*size*size;
int const weights_pre_index = fil*new_lda + chan*size*size;
int const input_pre_index = chan*in_w*in_h;

// filter - y
for (f_y = 0; f_y < size; ++f_y)
{
int input_y = y + f_y - pad;
// filter - x
for (f_x = 0; f_x < size; ++f_x)
{
int input_x = x + f_x - pad;
if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

int input_index = input_pre_index + input_y*in_w + input_x;
int weights_index = weights_pre_index + f_y*size + f_x;
//int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
//int weights_index = fil*new_lda + chan*size*size + f_y*size + f_x;

uint8_t in_bit = get_bit((uint8_t *)input, input_index);
uint8_t w_bit = get_bit((uint8_t *)weights, weights_index);
int res = xnor_bit1(in_bit, w_bit);
sum += res;
good_val++;

//sum += input[input_index] *weights[weights_index];

}
}
// l.output[filters][width][height] +=
//        state.input[channels][width][height] *
//        l.weights[filters][channels][filter_width][filter_height];
//output[output_index] += sum;
}
sum = sum - (good_val - sum);
output[output_index] = sum * mean_arr_gpu[fil]; // atoimcAdd for inter-BLOCK sum
}

}