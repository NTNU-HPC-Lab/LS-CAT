#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *scale, int batch, int filters, int spatial, int current_size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= current_size) return;

int f = (index / spatial) % filters;
output[index] *= scale[f];
}