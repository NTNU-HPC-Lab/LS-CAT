#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_add_kernel(int N, float ALPHA, float BETA, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < N) X[i*INCX] = X[i*INCX] * ALPHA + BETA;
}