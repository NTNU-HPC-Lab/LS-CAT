#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_weight_updates_kernel(int N, float coef, float *weights_gpu, float *weight_updates_gpu)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < N) {
const float w = weights_gpu[i];
const float wu = weight_updates_gpu[i];
const float wu_sign = (wu == 0) ? 0 : (fabs(wu) / wu);
const float abs_limit = fabs(w * coef);
if (fabs(wu) > abs_limit) weight_updates_gpu[i] = abs_limit * wu_sign;
}
}