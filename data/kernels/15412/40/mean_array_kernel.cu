#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_array_kernel(float *src, int size, float alpha, float *avg)
{
const int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i >= size) return;

avg[i] = avg[i] * (1 - alpha) + src[i] * alpha;
src[i] = avg[i];
}