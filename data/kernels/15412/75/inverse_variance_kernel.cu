#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inverse_variance_kernel(int size, float *src, float *dst, float epsilon)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size)
dst[index] = 1.0f / sqrtf(src[index] + epsilon);
}