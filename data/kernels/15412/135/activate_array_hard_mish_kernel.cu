#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float hard_mish_yashas(float x)
{
if (x > 0)
return x;
if (x > -2)
return x * x / 2 + x;
return 0;
}
__device__ float mish_yashas(float x)
{
float e = __expf(x);
if (x <= -18.0f)
return x * e;

float n = e * e + 2 * e;
if (x <= -5.0f)
return x * __fdividef(n, n + 2);

return x - 2 * __fdividef(x, n + 2);
}
__global__ void activate_array_hard_mish_kernel(float *x, int n, float *activation_input, float *output_gpu)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {

float x_val = x[i];
if (activation_input) activation_input[i] = x_val;    // store value before activation
output_gpu[i] = hard_mish_yashas(x_val);
}
}