#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reset_nan_and_inf_kernel(float *input, size_t size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
float val = input[index];
if (isnan(val) || isinf(val)) {
input[index] = 0;
}
}
}