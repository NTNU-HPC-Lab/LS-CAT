#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, float *scales_c, float *out)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
if (scale_wh) {
int osd_index = index % channel_size + (index / batch_size)*channel_size;

out[index] = in_w_h_c[index] * scales_c[osd_index];
}
else {
out[index] = in_w_h_c[index] * scales_c[index / channel_size];
}
}
}