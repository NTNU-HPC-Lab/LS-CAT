#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "sam_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *in_w_h_c = NULL;
hipMalloc(&in_w_h_c, XSIZE*YSIZE);
int size = XSIZE*YSIZE;
int channel_size = XSIZE*YSIZE;
float *scales_c = NULL;
hipMalloc(&scales_c, XSIZE*YSIZE);
float *out = NULL;
hipMalloc(&out, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c,size,channel_size,scales_c,out);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c,size,channel_size,scales_c,out);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c,size,channel_size,scales_c,out);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}