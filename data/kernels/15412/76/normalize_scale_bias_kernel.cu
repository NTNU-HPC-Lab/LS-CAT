#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_scale_bias_kernel(int N, float *x, float *mean, float *variance, float *scales, float *biases, int batch, int filters, int spatial, int inverse_variance, float epsilon)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= N) return;
int f = (index / spatial) % filters;

float val = 0;
if(inverse_variance) val = (x[index] - mean[f]) * variance[f];
else val = (x[index] - mean[f]) / (sqrtf(variance[f] + epsilon));
val *= scales[f];
val += biases[f];

if (!isnan(val) && !isinf(val))
x[index] = val;
}