#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_zero_nonmax_kernel(int n, float *input, float *output)
{

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) return;

if (input[id] != output[id]) output[id] = 0;
}