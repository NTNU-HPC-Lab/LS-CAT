#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void activate_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *output_gpu)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

int wh_i = i % wh_step;
int b = i / wh_step;

const float eps = 0.0001;
if (i < size) {
float sum = eps;
int k;
for (k = 0; k < channels; ++k) {
float val = x[wh_i + k * wh_step + b*wh_step*channels];
if (val > 0) sum += val;
}
for (k = 0; k < channels; ++k) {
float val = x[wh_i + k * wh_step + b*wh_step*channels];
if (val > 0) val = val / sum;
else val = 0;
output_gpu[wh_i + k * wh_step + b*wh_step*channels] = val;
}
}
}