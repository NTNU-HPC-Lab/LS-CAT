#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int batch, float *delta, float *prev_delta, int *indexes)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) return;

int index = indexes[id];
prev_delta[index] += delta[id];
}