#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_scales_dropblock_kernel(float *drop_blocks_scale, int block_size_w, int block_size_h, int outputs, int batch)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index >= batch) return;

//printf(" drop_blocks_scale[index] = %f \n", drop_blocks_scale[index]);
const float prob = drop_blocks_scale[index] / (float)outputs;
const float scale = 1.0f / (1.0f - prob);
drop_blocks_scale[index] = scale;
}