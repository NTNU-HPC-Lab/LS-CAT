#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
float diff = truth[i] - pred[i];
float abs_val = abs(diff);
if(abs_val < 1) {
error[i] = diff * diff;
delta[i] = diff;
}
else {
error[i] = 2*abs_val - 1;
delta[i] = (diff < 0) ? -1 : 1;
}
}
}