#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREADS 512


__global__ void opt_cond_itr(int num_train_cases, double *opt_cond, double alpha_high, double alpha_high_prev, int high_label, int high_indx, double alpha_low, double alpha_low_prev, int low_label, int low_indx, double *kernel_val_mat){

int global_id = blockIdx.x * blockDim.x + threadIdx.x;

if(global_id < num_train_cases){
opt_cond[global_id] += (alpha_high - alpha_high_prev) * high_label * kernel_val_mat[high_indx*num_train_cases+global_id]
+ (alpha_low - alpha_low_prev) * low_label * kernel_val_mat[low_indx*num_train_cases+global_id];
}
}