#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "opt_cond_itr.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int num_train_cases = 1;
double *opt_cond = NULL;
hipMalloc(&opt_cond, XSIZE*YSIZE);
double alpha_high = 2;
double alpha_high_prev = 2;
int high_label = 1;
int high_indx = 1;
double alpha_low = 2;
double alpha_low_prev = 2;
int low_label = 1;
int low_indx = 1;
double *kernel_val_mat = NULL;
hipMalloc(&kernel_val_mat, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
opt_cond_itr<<<gridBlock,threadBlock>>>(num_train_cases,opt_cond,alpha_high,alpha_high_prev,high_label,high_indx,alpha_low,alpha_low_prev,low_label,low_indx,kernel_val_mat);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
opt_cond_itr<<<gridBlock,threadBlock>>>(num_train_cases,opt_cond,alpha_high,alpha_high_prev,high_label,high_indx,alpha_low,alpha_low_prev,low_label,low_indx,kernel_val_mat);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
opt_cond_itr<<<gridBlock,threadBlock>>>(num_train_cases,opt_cond,alpha_high,alpha_high_prev,high_label,high_indx,alpha_low,alpha_low_prev,low_label,low_indx,kernel_val_mat);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}