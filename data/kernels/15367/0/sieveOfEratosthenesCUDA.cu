#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void sieveOfEratosthenesCUDA(char *primes, uint64_t max)

{

// first thread 0
if (threadIdx.x == 0 && threadIdx.y == 0)   {

primes[0] = 1; // value of 1 means the number is NOT prime
primes[1] = 1; // numbers "0" and "1" are not prime numbers

// sieve multiples of two

for (int j = 2 * 2; j < max; j += 2)      {

primes[j] = 1;

}

}

else

{

int index = blockIdx.x * blockDim.x + threadIdx.x;

const uint64_t maxRoot = sqrt((double)max);



// make sure index won't go out of bounds, also don't execute it

// on index 1

if (index < maxRoot && primes[index] == 0 && index > 1)

{

// mark off the composite numbers

for (int j = index * index; j < max; j += index)

{

primes[j] = 1;

}

}

}

}