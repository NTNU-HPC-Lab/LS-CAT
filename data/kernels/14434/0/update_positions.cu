#include "hip/hip_runtime.h"
#include "includes.h"

float * g_outputs_d, *g_sweepers_d_2;

__global__ void update_positions(float max_speed, float * outputs_d, float * sweepers_d)
{
int my_index = blockIdx.x * blockDim.x + threadIdx.x;

sweepers_d[my_index] +=  (2 * outputs_d[my_index] * max_speed) - max_speed;
}