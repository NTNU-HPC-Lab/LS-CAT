#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_distances(float * sweeper_pos_v, float * mine_pos_v, int num_sweepers, int num_mines, float * distance_v, float * inputs, int * sweeper_score_v, int width, int height, int size)
{
#define sweeperIdx blockIdx.y
#define mineIdx threadIdx.x*2

int distanceIdx = (blockIdx.y * num_mines) + threadIdx.x;
float vec_x;
float vec_y;
float distance;

__shared__ float sweeper_pos[2];

if (threadIdx.x < 2)
{
sweeper_pos[threadIdx.x] = sweeper_pos_v[sweeperIdx + threadIdx.x];
inputs[((sweeperIdx * 4) + threadIdx.x) + 2] = sweeper_pos[threadIdx.x]; //copy the sweeper position out to the inputs for the neural network in parallel

}

__syncthreads();


vec_x = mine_pos_v[mineIdx] - sweeper_pos[0];
vec_y = mine_pos_v[mineIdx + 1] - sweeper_pos[1];
distance = sqrt((vec_x * vec_x) + (vec_y * vec_y));
distance_v[distanceIdx] = distance;

if (distance < size)
{
/*
mine_pos_v[mineIdx] = width / 2;
mine_pos_v[mineIdx + 1] = height / 2;
*/

mine_pos_v[mineIdx] = ((threadIdx.x + 1 ) * clock()) % width;
mine_pos_v[mineIdx + 1] = ((threadIdx.x + 1) * clock()) % height;


sweeper_score_v[sweeperIdx]++;
}

#undef sweeperIdx
#undef mineIdx
}