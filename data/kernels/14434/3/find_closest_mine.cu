#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_closest_mine(float * mine_pos_v, float * distances_v, int * mineIdx_v, int num_sweeprs, int num_mines, float * inputs)
{
#define sweeperIdx blockIdx.y
#define first_item blockIdx.y*num_mines
int my_index = (gridDim.x * blockIdx.x) + threadIdx.x;

//mineIdx_v[sweeperIdx * num_mines + threadIdx.x] = threadIdx.x;
mineIdx_v[sweeperIdx * num_mines + my_index] = my_index;

for (int stride = num_mines / 2; stride > 1; stride /= 2)
{
__syncthreads();
if (my_index < stride)
{
if (distances_v[my_index + first_item] < distances_v[my_index + first_item + stride])
{
distances_v[my_index + first_item] = distances_v[my_index + first_item + stride];
mineIdx_v[my_index + first_item] = mineIdx_v[my_index + first_item + stride];
}
}
}

inputs[sweeperIdx * 4] = mine_pos_v[mineIdx_v[sweeperIdx] * 2];
inputs[sweeperIdx * 4 + 1] = mine_pos_v[mineIdx_v[sweeperIdx] * 2 + 1];

#undef sweeperIdx
#undef first_item
}