#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_neural_net(float *Weights_D, int num_per_sweeper, int num_per_layer, int num_per_input, int num_per_output, int num_weights, int num_layers, float response, float *inputs_d, float *outputs_d)
{

extern __shared__ float buffer[];

int start_of_weights = blockIdx.x * num_weights;
int start_of_hidden_layers = start_of_weights + (num_per_input * num_per_layer);


//input layer
buffer[threadIdx.x] = 0;
for (int i = 0; i < num_per_input; ++i)
{
buffer[threadIdx.x] += inputs_d[(blockIdx.x * num_per_input) + i] * Weights_D[start_of_weights + (threadIdx.x * num_per_input) + i];
}
buffer[threadIdx.x] = 1.0 / (1.0 + exp(-buffer[threadIdx.x] / response));
__syncthreads();

//subsequent hidden layers
float temp;

for (int i = 0; i < num_layers; ++i)
{
temp = 0;
for (int j = 0; j < num_per_layer; ++j)
{
temp += buffer[j] * Weights_D[start_of_hidden_layers + (num_per_layer * num_per_layer * i) + (num_per_layer * threadIdx.x) + j];
}
temp = 1.0 / (1.0 + exp(-temp / response));

__syncthreads();
buffer[threadIdx.x] = temp;
__syncthreads();
}

//output layer
if (threadIdx.x < num_per_output)
{
temp = 0;
for (int i = 0; i < num_per_layer; ++i)
{
temp += buffer[i] * Weights_D[start_of_hidden_layers + (num_per_layer * num_per_layer * num_layers) + (num_per_layer * threadIdx.x) + i];
}
temp = 1.0 / (1.0 + exp(-temp / response));

__syncthreads();

//copy the result back out to the outputs vector
outputs_d[(blockIdx.x * num_per_output) + threadIdx.x] = temp;
}

}