#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_lut(int *lut, int * hist_in, int img_size, int nbr_bin){


__shared__ int shared_hist[256];
shared_hist[threadIdx.x] = hist_in[threadIdx.x];
__syncthreads();
__shared__  int cdf[256];
__syncthreads();

int i, min, d;
//int cdf = 0;
min = 0;
i = 0;

while(min == 0){
min = shared_hist[i++];
}
d = img_size - min;
for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
__syncthreads();
shared_hist[threadIdx.x] += shared_hist[threadIdx.x-stride];
}
cdf[threadIdx.x] = shared_hist[threadIdx.x];
//printf("cdf = %d\n",cdf);
__syncthreads();



//for(i = 0; i <= threadIdx.x; i ++){	//tha mporouse na ginei me prefix sum san veltistoipohsh FIXME
//  cdf += shared_hist[i];
//  lut[i] = (cdf - min)*(nbr_bin - 1)/d;
//}
//printf("cdf = %d\n",cdf);


lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
if(lut[threadIdx.x] < 0){
lut[threadIdx.x] = 0;
}
}