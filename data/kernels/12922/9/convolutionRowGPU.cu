#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionRowGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){

int k;
double sum = 0;
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
for (k = -filterR; k <= filterR; k++) {
int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
if (d >= 0 && d < imageW) {
sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
}
h_Dst[iy * imageW + ix] = sum;
}
}