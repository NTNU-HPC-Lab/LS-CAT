#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_lut(int *lut, int * hist_in, int img_size, int nbr_bin){


__shared__ int shared_hist[256];
shared_hist[threadIdx.x] = hist_in[threadIdx.x];
__syncthreads();

int i, cdf, min, d;
cdf = 0;
min = 0;
i = 0;

while(min == 0){
min = shared_hist[i++];
}
d = img_size - min;
for(i = 0; i <= threadIdx.x; i ++){	//tha mporouse na ginei me prefix sum san veltistoipohsh FIXME
cdf += shared_hist[i];
//lut[i] = (cdf - min)*(nbr_bin - 1)/d;
}

lut[threadIdx.x] = (int)(((float)cdf - min)*255/d + 0.5);
if(lut[threadIdx.x] < 0){
lut[threadIdx.x] = 0;
}
}