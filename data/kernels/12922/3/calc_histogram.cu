#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int glb_hist[COLORS];   __global__ void calc_histogram(unsigned char * img_in, int offset_start, int offset_end){
int ix = blockIdx.x * blockDim.x + threadIdx.x;;
const int gridW = gridDim.x * blockDim.x;
int Row, pos;

__shared__ int hist[COLORS];

if (threadIdx.x < COLORS) {
hist[threadIdx.x] = 0;
}
__syncthreads();

int RowNum = (offset_end - offset_start) / gridW;
int extras = (offset_end - offset_start) % gridW;



for (Row=0; Row<RowNum; Row++) {
pos = Row * gridW + ix;
atomicAdd(&hist[img_in[pos + offset_start]],1);
}
if (extras && ix < extras) {
pos = Row * gridW + ix;
atomicAdd(&hist[img_in[pos + offset_start]],1);
}
__syncthreads();
if (threadIdx.x < COLORS) {
atomicAdd(&glb_hist[threadIdx.x],hist[threadIdx.x]);
}
}
__global__ void calc_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){

int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
const int gridW = gridDim.x * blockDim.x;
int img_position = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW

// __shared__ hist_out[nbr_bin];

if (img_position < nbr_bin) { //allagh tou img_position me threadIdx.x se auto to if anevazei poly to contrast
hist_out[img_position] = 0;
}

__syncthreads();

if(img_position < img_size){
atomicAdd(&hist_out[img_in[img_position]],1);
}
__syncthreads();
}