#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
const int gridW = gridDim.x * blockDim.x;
int img_position1 = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW
__syncthreads();
if(img_position1 < img_size){
if(lut[img_in[img_position1]] > 255){
img_out[img_position1] = 255;
}
else{
img_out[img_position1] = (unsigned char)lut[img_in[img_position1]];
}
}


}