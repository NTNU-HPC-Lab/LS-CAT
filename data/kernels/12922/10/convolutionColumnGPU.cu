#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionColumnGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){
int k;
double sum = 0;
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
for (k = -filterR; k <= filterR; k++) {
int d = iy + k;
if (d >= 0 && d < imageH) {
sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
}
h_Dst[iy * imageW + ix] = sum;
}
}