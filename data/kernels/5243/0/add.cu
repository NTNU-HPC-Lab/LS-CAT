#include "hip/hip_runtime.h"
#include "includes.h"


// CUDA kernel to add elements

__global__    void add(int N, float *x)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i<N)
x[i] = x[i] *2;
}