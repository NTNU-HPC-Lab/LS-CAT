#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(int *A, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
A[i] = A[i] * 2;
}
}