#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *fData, int *sData, int *oData, int x, int y){

int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for(int i = index; i < x*y; i += stride)
{
oData[i] = fData[i] + sData[i];
}
}