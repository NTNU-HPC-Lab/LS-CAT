#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convn_valid_kernel(float *output, float *data, float *kernel, const int H, const int W, const int kH, const int kW) {

// Matrix index
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// vH, vW stands for valid H and valid W
const int vH = H - kH + 1;
const int vW = W - kW + 1;

if (x >= vH || y >= vW)
return;

x += kH - 1;
y += kW - 1;

float sum = 0;
for (int i = 0; i < kW; ++i)
for(int j = 0; j < kH; ++j)
sum += kernel[ i * kH + j ] * data[ (y - i) * H + (x - j) ];

x -= kH - 1;
y -= kW - 1;

output[ y * vH + x ] = sum;
}