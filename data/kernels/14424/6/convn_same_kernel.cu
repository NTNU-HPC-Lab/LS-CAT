#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convn_same_kernel(float *output, float *data, float *kernel, const int H, const int W, const int kH, const int kW) {

// Matrix index
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x >= H || y >= W)
return;

const int i0 = kW / 2, j0 = kH / 2;

float sum = 0;
for (int i = 0; i < kW; ++i) {
for(int j = 0; j < kH; ++j) {
int ii = y - i + i0;
int jj = x - j + j0;

if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
continue;

sum += kernel[ i * kH + j ] * data[ ii * H + jj ];
}
}

output[y * H + x] = sum;
}