#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convn_full_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) {

// Matrix index
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// fH, fW stands for full H and full W
const int fH = H + kH - 1;
const int fW = W + kW - 1;

if (x >= fH || y >= fW)
return;

float sum = 0;
for (int i = 0; i < kW; ++i) {
for(int j = 0; j < kH; ++j) {
int ii = y - i;
int jj = x - j;

if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
continue;

sum += kernel[ i * kH + j ] * data[ ii * H + jj ];
}
}

output[ y * fH + x ] = sum;
}