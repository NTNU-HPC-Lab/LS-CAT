#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
float cur_weight = *cur_tot_weight;

float new_num_frames = old_num_frames + cur_weight;
float prior_scale_change = 1.0f;

if(max_count!=0.0f) {
float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
prior_scale_change += new_prior_scale - old_prior_scale;
}

for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
i += blockDim.x * gridDim.x) {
int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
quadratic[diag_idx] += prior_scale_change;
}

if (threadIdx.x == 0 && blockIdx.x==0) {
linear[0] += prior_offset * prior_scale_change;
}
}