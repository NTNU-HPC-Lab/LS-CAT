#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32














__global__ void binary_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

if(x == y && x < n_a && threadIdx.x == 0) {
d[y * pitch_d + x] = 0.0;
}

// If all element is to be computed
if(y < n_a && x < y) {
__shared__ float temp[2 * NUM_THREADS];

temp[threadIdx.x] = 0.0;
temp[threadIdx.x + NUM_THREADS] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
int a = vg_a[x * pitch_a + offset] != 0.0;
int b = vg_a[y * pitch_a + offset] != 0.0;
if(a ^ b) {
temp[threadIdx.x] += 1.0;
}
if(a || b) {
temp[threadIdx.x + NUM_THREADS] += 1.0;
}
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
temp[threadIdx.x + NUM_THREADS] += temp[threadIdx.x + stride + NUM_THREADS];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
float val = temp[0];
if(temp[NUM_THREADS] != 0.0) {
val /= temp[NUM_THREADS];
}
d[y * pitch_d + x] = val;
d[x * pitch_d + y] = val;
}
}
}