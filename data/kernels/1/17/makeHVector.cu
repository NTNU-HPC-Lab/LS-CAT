#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512





// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void makeHVector(int rows, float * input, float * output)
{
int
i, j;
float
elt, sum;
__shared__ float
beta, sums[NTHREADS];

if(threadIdx.x >= rows)
return;

sum = 0.f;
for(i = threadIdx.x ; i < rows; i += NTHREADS) {
if((threadIdx.x == 0) && (i == 0))
continue;
elt = input[i];
output[i] = elt;
sum += elt * elt;
}
sums[threadIdx.x] = sum;
__syncthreads();

for(i = blockDim.x >> 1; i > 0 ; i >>= 1) {
j = i+threadIdx.x;
if((threadIdx.x < i) && (j < rows))
sums[threadIdx.x] += sums[j];
__syncthreads();
}

if(threadIdx.x == 0) {
elt = input[0];
float norm = sqrtf(elt * elt + sums[0]);

if(elt > 0)
elt += norm;
else
elt -= norm;

output[0] = elt;

norm = elt * elt + sums[0];
beta = sqrtf(2.f / norm);
}
__syncthreads();

for(i = threadIdx.x; i < rows; i += NTHREADS)
output[i] *= beta;
}