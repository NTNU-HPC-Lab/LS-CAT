#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale(float knot_max, int nx, int nsamples, float * x, int pitch_x)
{
int
col_idx = blockDim.x * blockIdx.x + threadIdx.x;

if(col_idx >= nx) return;

float
min, max,
* col = x + col_idx * pitch_x;

// find the min and the max
min = max = col[0];
for(int i = 1; i < nsamples; i++) {
if(col[i] < min) min = col[i];
if(col[i] > max) max = col[i];
}

float delta = max - min;
for(int i = 0; i < nsamples; i++)
col[i] = (knot_max * (col[i] - min)) / delta;
}