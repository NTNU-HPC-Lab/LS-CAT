#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32














__global__ void dot_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

// If all element is to be computed
if(y < n_a && x <= y) {
__shared__ float temp[NUM_THREADS];

temp[threadIdx.x] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
float t = vg_a[x * pitch_a + offset] * vg_a[y * pitch_a + offset];
temp[threadIdx.x] += t;
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
d[y * pitch_d + x] = temp[0];
d[x * pitch_d + y] = temp[0];
}
}
}