#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32














__global__ void maximum_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t
x = blockIdx.x, y = blockIdx.y;

if(x == y && x < n_a && threadIdx.x == 0) {
d[y * pitch_d + x] = 0.0;
}

// If all element is to be computed
if(y < n_a && x < y) {
__shared__ float temp[NUM_THREADS];

temp[threadIdx.x] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
float t = abs(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
temp[threadIdx.x] = max(t, temp[threadIdx.x]);
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] = max(temp[threadIdx.x], temp[threadIdx.x + stride]);
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
float s = temp[0];
d[y * pitch_d + x] = s;
d[x * pitch_d + y] = s;
}
}
}