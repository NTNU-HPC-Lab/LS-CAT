#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__global__ void gpuSD(const float * vectsA, size_t na, const float * vectsB, size_t nb, size_t dim, const float * means, const float * numPairs, float * sds)
{
size_t
offset, stride,
tx = threadIdx.x,
bx = blockIdx.x, by = blockIdx.y;
float
a, b,
termA, termB;
__shared__ float
meanA, meanB, n,
threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

if((bx >= na) || (by >= nb))
return;

if(tx == 0) {
meanA = means[bx*nb*2+by*2];
meanB = means[bx*nb*2+by*2+1];
n = numPairs[bx*nb+by];
}
__syncthreads();

threadSumsA[tx] = 0.f;
threadSumsB[tx] = 0.f;
for(offset = tx; offset < dim; offset += NUMTHREADS) {
a = vectsA[bx * dim + offset];
b = vectsB[by * dim + offset];
if(!(isnan(a) || isnan(b))) {
termA = a - meanA;
termB = b - meanB;
threadSumsA[tx] += termA * termA;
threadSumsB[tx] += termB * termB;
}
}
__syncthreads();

for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
if(tx < stride) {
threadSumsA[tx] += threadSumsA[tx + stride];
threadSumsB[tx] += threadSumsB[tx + stride];
}
__syncthreads();
}
if(tx == 0) {
sds[bx*nb*2+by*2]   = sqrtf(threadSumsA[0] / (n - 1.f));
sds[bx*nb*2+by*2+1] = sqrtf(threadSumsB[0] / (n - 1.f));
}
}