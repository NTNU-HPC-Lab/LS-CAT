#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__global__ void noNAsPmccMeans(int nRows, int nCols, float * a, float * means)
{
int
col = blockDim.x * blockIdx.x + threadIdx.x,
inOffset = col * nRows,
outOffset = threadIdx.x * blockDim.y,
j = outOffset + threadIdx.y;
float sum = 0.f;

if(col >= nCols) return;

__shared__ float threadSums[NUMTHREADS*NUMTHREADS];

for(int i = threadIdx.y; i < nRows; i += blockDim.y)
sum += a[inOffset + i];

threadSums[j] = sum;
__syncthreads();

for(int i = blockDim.y >> 1; i > 0; i >>= 1) {
if(threadIdx.y < i) {
threadSums[outOffset+threadIdx.y]
+= threadSums[outOffset+threadIdx.y + i];
}
__syncthreads();
}
if(threadIdx.y == 0)
means[col] = threadSums[outOffset] / (float)nRows;
}