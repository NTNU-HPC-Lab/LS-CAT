#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512





// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void getColNorms(int rows, int cols, float * da, int lda, float * colNorms)
{
int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
float
sum = 0.f, term,
* col;

if(colIndex >= cols)
return;

col = da + colIndex * lda;

// debug printing
// printf("printing column %d\n", colIndex);
// for(int i = 0; i < rows; i++)
// printf("%f, ", col[i]);
// puts("");
// end debug printing

for(int i = 0; i < rows; i++) {
term = col[i];
term *= term;
sum += term;
}

// debug printing
// printf("norm %f\n", norm);
// end debug printing

colNorms[colIndex] = sum;
}