#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32














__global__ void binary_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

// If all element is to be computed
if(x < n_a && y < n_b) {
__shared__ float temp[2 * NUM_THREADS];

temp[threadIdx.x] = 0.0;
temp[threadIdx.x + NUM_THREADS] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
int a = vg_a[x * pitch_a + offset] != 0.0;
int b = vg_b[y * pitch_b + offset] != 0.0;
if(a ^ b) {
temp[threadIdx.x] += 1.0;
}
if(a || b) {
temp[threadIdx.x + NUM_THREADS] += 1.0;
}
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
temp[threadIdx.x + NUM_THREADS] += temp[threadIdx.x + stride + NUM_THREADS];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
float val = temp[0];
if(temp[NUM_THREADS] != 0.0) {
val /= temp[NUM_THREADS];
}
d[y * pitch_d + x] = val;
}
}
}