#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__global__ void gpuMeansNoTest(const float * vectsA, size_t na, const float * vectsB, size_t nb, size_t dim, float * means, float * numPairs)
{
size_t
offset, stride,
bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
float a, b;

__shared__ float
threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS],
count[NUMTHREADS];

if((bx >= na) || (by >= nb))
return;

threadSumsA[tx] = 0.f;
threadSumsB[tx] = 0.f;
count[tx] = 0.f;

for(offset = tx; offset < dim; offset += NUMTHREADS) {
a = vectsA[bx * dim + offset];
b = vectsB[by * dim + offset];

threadSumsA[tx] += a;
threadSumsB[tx] += b;
count[tx] += 1.f;
}
__syncthreads();

for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
if(tx < stride) {
threadSumsA[tx] += threadSumsA[tx + stride];
threadSumsB[tx] += threadSumsB[tx + stride];
count[tx] += count[tx+stride];
}
__syncthreads();
}
if(tx == 0) {
means[bx*nb*2+by*2] = threadSumsA[0] / count[0];
means[bx*nb*2+by*2+1] = threadSumsB[0] / count[0];
numPairs[bx*nb+by] = count[0];
}
}