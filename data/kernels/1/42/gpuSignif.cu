#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__global__ void gpuSignif(const float * gpuNumPairs, const float * gpuCorrelations, size_t n, float * gpuTScores)
{
size_t
i, start,
bx = blockIdx.x, tx = threadIdx.x;
float
radicand, cor, npairs;

start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;
for(i = 0; i < THREADWORK; i++) {
if(start+i >= n)
break;

npairs = gpuNumPairs[start+i];
cor = gpuCorrelations[start+i];
radicand = (npairs - 2.f) / (1.f - cor * cor);
gpuTScores[start+i] = cor * sqrtf(radicand);
}
}