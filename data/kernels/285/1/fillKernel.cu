#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillKernel(int *a, int n) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < n) a[tid] = tid;
}