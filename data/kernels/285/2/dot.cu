#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float * val, int *row_ind, int *col_ind, int nnz, float * ret, float * w) {
const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < nnz) {
int r = row_ind[tid];
int c = col_ind[tid];
float v = val[tid];
atomicAdd(&ret[r], v * w[c]);
}
}