#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grad(float * val, int * row_ind, int *col_ind, float * mat_err, int nnz, float *act, float *label, float *w, float learning_rate) {
const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < nnz) {
int r = row_ind[tid];
int c = col_ind[tid];
float v = val[tid];
mat_err[tid] = abs(label[r] - act[r]);
float err = v * (label[r] - act[r]);
atomicAdd(&w[c], learning_rate * err);
}
}