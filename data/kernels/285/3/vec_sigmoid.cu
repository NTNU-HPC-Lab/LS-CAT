#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vec_sigmoid(float * d, int num) {
const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < num) {
if(d[tid] > 10.0) d[tid] = 1.0;
else if(d[tid] < -10.0) d[tid] = 0.0;
else d[tid] = 1.0 / (1.0 + exp(-1.0 * d[tid]));
}
}