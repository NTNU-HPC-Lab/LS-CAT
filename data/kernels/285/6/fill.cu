#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill(float * w, float val, int size) {
const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < size) w[tid] = val;
}