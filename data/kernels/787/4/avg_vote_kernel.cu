#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void avg_vote_kernel( float *A, float *B, int *corrAB, int patch, int c, int h, int w )
{
int _id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
int radius = patch / 2;
if (_id < c * size) {
// _id = dc * size + id
int id = _id % size, dc = _id / size;
int x1 = id % w, y1 = id / w;
double sum = 0;
int    cnt = 0;
for (int dx = -radius; dx <= radius; dx++) {
for (int dy = -radius; dy <= radius; dy++) {
int new_x1 = x1 + dx, new_y1 = y1 + dy;

if (new_x1 >= 0 && new_x1 < w && new_y1 >= 0 && new_y1 < h) {
int new_id1 = new_y1 * w + new_x1;
int x2 = corrAB[new_id1 * 2 + 0];
int y2 = corrAB[new_id1 * 2 + 1];
int new_x2 = x2 - dx, new_y2 = y2 - dy;

if (new_x2 >= 0 && new_x2 < w && new_y2 >= 0 && new_y2 < h) {
int new_id2 = new_y2 * w + new_x2;
sum += A[dc * size + new_id2];
cnt++;
}
}
}
}
if (cnt != 0)
B[dc * size + id] = sum / cnt;

}
return ;
}