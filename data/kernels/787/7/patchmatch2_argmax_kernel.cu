#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void patchmatch2_argmax_kernel( float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, int c, int h, int w )
{
int h1 = h, h2 = h, w1 = w, w2 = w;
int id1 = blockIdx.x * blockDim.x + threadIdx.x;
int size1 = h1 * w1;//, size2 = h2 * w2;
int s_size = 2 * s_rad + 1;
int s_n = s_size * s_size;

if (id1 < size1) {
float conv_max = -1;

// int y1 = id1 / w1, x1 = id1 % w1;

int x2 = prev_corrAB_upsampled[2 * id1 + 0];
int y2 = prev_corrAB_upsampled[2 * id1 + 1];

for (int dx2 = -s_rad; dx2 <= s_rad; dx2++) {
for (int dy2 = -s_rad; dy2 <= s_rad; dy2++) {
int new_y2 = y2 + dy2;
int new_x2 = x2 + dx2;

if (new_x2 >= 0 && new_x2 < w2 && new_y2 >= 0 && new_y2 < h2) {
int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
int id = id1 * s_n + s_idx;
float conv_result = conv[id];
if (conv_result > conv_max) {
conv_max = conv_result;
corrAB[id1 * 2 + 0] = new_x2;
corrAB[id1 * 2 + 1] = new_y2;
}
}
}
}
}

return ;
}