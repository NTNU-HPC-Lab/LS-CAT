#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void patchmatch2_conv_kernel( float *A, float *B, float *AP, float *BP, float *conv, int *prev_corrAB_upsampled, int patch, int s_rad, int c, int h, int w )
{
int h1 = h, h2 = h, w1 = w, w2 = w;
int _id = blockIdx.x * blockDim.x + threadIdx.x;
int size1 = h * w, size2 = h * w;
int s_size = 2 * s_rad + 1;
int s_n = s_size * s_size;
if (_id < size1 * s_n) {
conv[_id] = -1;

int id1 = _id / s_n, s_idx = _id % s_n;
int y1 = id1 / w1, x1 = id1 % w1;
int dy2 = s_idx / s_size - s_rad, dx2 = s_idx % s_size - s_rad;

int x2 = prev_corrAB_upsampled[2 * id1 + 0];
int y2 = prev_corrAB_upsampled[2 * id1 + 1];

int new_y2 = y2 + dy2;
int new_x2 = x2 + dx2;
if (!(new_x2 >= 0 && new_x2 < w2 && new_y2 >= 0 && new_y2 < h2)) {
return ;
}

// Improve by local searching
int kernel_radius = (patch - 1) / 2;
float conv_result = 0;
int cnt = 0;
for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
int xx1 = x1 + dx, yy1 = y1 + dy;
int xx2 = new_x2 + dx, yy2 = new_y2 + dy;
if (0 <= xx1 && xx1 < w1 && 0 <= yy1 && yy1 < h1 &&
0 <= xx2 && xx2 < w2 && 0 <= yy2 && yy2 < h2)
{
int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w2 + xx2;
for (int dc = 0; dc < c; dc++) {
float term1 = A[dc * size1 + _id1];
float term2 = B[dc * size2 + _id2];
conv_result += term1 * term2;

term1 = AP[dc * size1 + _id1];
term2 = BP[dc * size2 + _id2];
conv_result += term1 * term2;
}
cnt++;

}
}
}

conv[_id] = conv_result / cnt;
}
return ;
}