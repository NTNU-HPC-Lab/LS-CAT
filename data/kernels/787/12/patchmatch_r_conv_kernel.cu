#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void patchmatch_r_conv_kernel( float *input, float *target, float *conv, int patch, int stride, int c1, int h1, int w1, int h2, int w2 )
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size1 = h1 * w1, size2 = h2 * w2;
int N = size1 * size2;
// id = id1 * size2 + id2

if (id < N) {
int id1 = id / size2, id2 = id % size2;

int x1 = id1 % w1, y1 = id1 / w1;
int x2 = id2 % w2, y2 = id2 / w2;

int kernel_radius = (patch - 1) / 2;

double conv_result = 0, norm_1 = 0, norm_2 = 0;
for (int dy = -kernel_radius; dy <= kernel_radius; dy+=stride) {
for (int dx = -kernel_radius; dx <= kernel_radius; dx+=stride) {
int xx1 = x1 + dx, yy1 = y1 + dy;
int xx2 = x2 + dx, yy2 = y2 + dy;
if (0 <= xx1 && xx1 < w1 && 0 <= yy1 && yy1 < h1 &&
0 <= xx2 && xx2 < w2 && 0 <= yy2 && yy2 < h2)
{
int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w2 + xx2;
for (int c = 0; c < c1; c++) {
float term1 = input[c * size1 + _id1];
float term2 = target[c * size2 + _id2];
conv_result += term1 * term2;
norm_1      += term1 * term1;
norm_2      += term2 * term2;
}

}
}
}

norm_1 = sqrt(norm_1);
norm_2 = sqrt(norm_2);

conv[id] = conv_result / (norm_1 * norm_2 + 1e-9);
}

return ;
}