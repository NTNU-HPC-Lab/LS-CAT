#include "hip/hip_runtime.h"
#include "includes.h"
//----------------------------------------------------------------------------------------------------------------------
/// @file CudaSPHKernals.cu
/// @author Declan Russell
/// @date 08/03/2015
/// @version 1.0
//----------------------------------------------------------------------------------------------------------------------

#define pi 3.14159265359f

//----------------------------------------------------------------------------------------------------------------------
/// @brief Kernal designed to produce a has key based on the location of a particle
/// @brief Hash function taken from Teschner, M., Heidelberger, B., Mueller, M., Pomeranets, D. and Gross, M.
/// @brief (2003). Optimized spatial hashing for collision detection of deformable objects
/// @param d_hashArray - pointer to a buffer to output our hash keys
/// @param d_posArray - pointer to the buffer that holds our particle positions
/// @param numParticles - the number of particles in our buffer
/// @param resolution - the resolution of our hash table
/// @param _gridScaler - Scales our points to between 0-1.
//----------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------
/// @brief This kernal is designed to count the cell occpancy of a hash table
/// @param d_hashArray - pointer to hash table buffer
/// @param d_cellOccArray - output array of cell occupancy count
/// @param _hashTableSize - the size of our hash table
/// @param _numPoints - the number of particles in our hashed array
//----------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our desity weighting kernal used in our navier stokes equations
/// @param _dst - the distance away of the neighbouring
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _densKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
//----------------------------------------------------------------------------------------------------------------------
__global__ void countCellOccKernal(unsigned int *d_hashArray, unsigned int *d_cellOccArray, int _hashTableSize, unsigned int _numPoints){
//Create our idx
int idx = threadIdx.x + blockIdx.x * blockDim.x;

// Make sure our idx is valid and add the occupancy count to the relevant cell
if ((idx < _numPoints) && (d_hashArray[idx] < _hashTableSize)) {
atomicAdd(&(d_cellOccArray[d_hashArray[idx]]), 1);
}
}