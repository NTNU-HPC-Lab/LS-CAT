#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillIntZeroKernal(int *_bufferPtr,int size)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx<size)
{
_bufferPtr[idx]=0;
}
}