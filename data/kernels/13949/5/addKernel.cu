#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *ic, const int *ia, const int *ib)
{
__syncthreads();
int i = threadIdx.x;
int b = blockIdx.x;
int bd = blockDim.x;
int gd = gridDim.x;

printf("G[%d] B[%d][%d]  t[%d]\n",gd,bd,b,i);
}