#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void GPUMult(int *A, int *B, int *C, int WIDTH)
{
int sol=0;
int i;i = threadIdx.x;
int j; j= threadIdx.y;

if (i < WIDTH && j < WIDTH) {
for (int k = 0; k < WIDTH; k++)
{
sol += A[j * WIDTH + k] * B[k * WIDTH + i];
}
C[j * WIDTH + i] = sol;
}

}