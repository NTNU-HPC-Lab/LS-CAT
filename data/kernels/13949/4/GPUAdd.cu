#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void GPUAdd(float *array1, float *array2, float *result, int WIDTH)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
result[i] = array1[i] + array2[i];
}