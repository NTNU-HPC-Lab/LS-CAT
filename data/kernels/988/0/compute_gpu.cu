#include "hip/hip_runtime.h"
#include "includes.h"
#define K 3
#define BLCH 8
#define BLCW 32

__global__ void compute_gpu(float *img, float *f, float * out, int bh, int bw, int imgH, int imgW, int imgN, int nF, int convH, int convW){
int idY = blockDim.y * blockIdx.y + threadIdx.y;
int idX = blockDim.x * blockIdx.x + threadIdx.x;

int inm1, inm2, inm3, inm4, inf, ind1, ind2, ind3;
inm1 = 0;
inf = 0;
ind1 = 0;

for (int mi = 0; mi < imgN; mi++){
ind1 += convW * convH;
inm1 += imgW * imgH;
if (idX < convH && idY < convW){
ind2 = ind1 + convW * idX;
inm2 = inm1 + imgW * idX;
ind3 = ind2 + idY;
inm3 = inm2 + idY;
for (int fi = 0; fi < nF; fi++){
inm4 = inm3 + imgW * fi;
inf = ind3*nF*nF + fi*nF;
for (int fj = 0; fj < nF; fj++){
out[ind3] += img[inm4+fj] * f[inf+fj];
}
}
}
}
}