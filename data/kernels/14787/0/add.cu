#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000
#define THREAD_PER_BLOCK 1

/**
* This macro checks return value of the CUDA runtime call and exits
* the application if the call failed.
*/
__global__ void add(int *a, int *b, int *c) {
int tid = blockIdx.x; // handle the data at this index
if (tid < N) {
c[tid] = a[tid] + b[tid];
}
}