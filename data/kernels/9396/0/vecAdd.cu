#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Maestría en Ciencias - Mención Informática
* -------------------------------------------
* Escriba un programa CUDA que calcule C = n*A + B, en donde A, B, C son vectores
* y n una constante escalar.
*
* Adaptado de https://www.olcf.ornl.gov/tutorials/cuda-vector-addition/
*
* Presentado por:
* Zuñiga Rojas, Gabriela
* Soncco Pimentel, Braulio
*/

hipEvent_t start, stop;
float elapsedTime;

const int k = 5;

// CUDA kernel. Each thread takes care of one element of c

__global__ void vecAdd(double *a, double *b, double *c, int n, int k)
{

// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
c[id] = k * a[id] + b[id];

}