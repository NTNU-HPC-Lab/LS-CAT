#include "hip/hip_runtime.h"
#include "includes.h"
const int listLength = 700;
__global__ void squareKernel(float* d_in, float *d_out, int threads_num) {
const unsigned int lid = threadIdx.x; // local id inside a block
const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
if (gid < threads_num){
d_out[gid] = powf((d_in[gid]/(d_in[gid]-2.3)),3);
}// do computation
}