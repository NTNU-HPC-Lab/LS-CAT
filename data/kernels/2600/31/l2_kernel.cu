#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
float diff = truth[i] - pred[i];
error[i] = diff * diff; //I know this is technically wrong, deal with it.
delta[i] = diff;
}
}