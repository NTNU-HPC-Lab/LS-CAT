#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) X[i*INCX] = ALPHA;
}