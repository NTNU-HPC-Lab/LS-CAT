#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}