#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;
int f = (index/spatial)%filters;

x[index] = (x[index] - mean[f])/(sqrt(variance[f]) + .000001f);
}