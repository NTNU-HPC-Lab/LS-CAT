#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}