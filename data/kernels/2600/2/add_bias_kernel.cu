#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
int offset = blockIdx.x * blockDim.x + threadIdx.x;
int filter = blockIdx.y;
int batch = blockIdx.z;

if(offset < size) output[(batch*n+filter)*size + offset] += biases[filter];
}