#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
float scale = 1./(batch * spatial);
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= filters) return;
int j,k;
mean[i] = 0;
for(j = 0; j < batch; ++j){
for(k = 0; k < spatial; ++k){
int index = j*filters*spatial + i*spatial + k;
mean[i] += x[index];
}
}
mean[i] *= scale;
}