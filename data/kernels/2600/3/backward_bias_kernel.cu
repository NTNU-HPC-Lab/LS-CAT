#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
__shared__ float part[BLOCK];
int i,b;
int filter = blockIdx.x;
int p = threadIdx.x;
float sum = 0;
for(b = 0; b < batch; ++b){
for(i = 0; i < size; i += BLOCK){
int index = p + i + size*(filter + n*b);
sum += (p+i < size) ? delta[index] : 0;
}
}
part[p] = sum;
__syncthreads();
if (p == 0) {
for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
}
}