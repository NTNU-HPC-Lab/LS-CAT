#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[i*INCY] *= X[i*INCX];
}