#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
const int threads = BLOCK;
__shared__ float local[threads];

int id = threadIdx.x;
local[id] = 0;

int filter = blockIdx.x;

int i, j;
for(j = 0; j < batch; ++j){
for(i = 0; i < spatial; i += threads){
int index = j*spatial*filters + filter*spatial + i + id;
local[id] += (i+id < spatial) ? delta[index] : 0;
}
}

if(id == 0){
mean_delta[filter] = 0;
for(i = 0; i < threads; ++i){
mean_delta[filter] += local[i];
}
mean_delta[filter] *= (-1./sqrt(variance[filter] + .000001f));
}
}