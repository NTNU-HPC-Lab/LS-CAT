#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
float scale = 1./(batch * spatial - 1);
int j,k;
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= filters) return;
variance[i] = 0;
for(j = 0; j < batch; ++j){
for(k = 0; k < spatial; ++k){
int index = j*filters*spatial + i*spatial + k;
variance[i] += pow((x[index] - mean[i]), 2);
}
}
variance[i] *= scale;
}