#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
int k;
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= groups) return;
sum[i] = 0;
for(k = 0; k < n; ++k){
sum[i] += x[k*groups + i];
}
}