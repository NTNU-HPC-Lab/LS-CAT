#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__device__ void softmax_device(int n, float *input, float temp, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
for(i = 0; i < n; ++i){
int val = input[i];
largest = (val>largest) ? val : largest;
}
for(i = 0; i < n; ++i){
float e = exp(input[i]/temp - largest/temp);
sum += e;
output[i] = e;
}
for(i = 0; i < n; ++i){
output[i] /= sum;
}
}
__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(b >= batch) return;
softmax_device(n, input + b*offset, temp, output + b*offset);
}