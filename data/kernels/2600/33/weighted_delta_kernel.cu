#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
if(da) da[i] += dc[i] * s[i];
db[i] += dc[i] * (1-s[i]);
ds[i] += dc[i] * a[i] + dc[i] * -b[i];
}
}