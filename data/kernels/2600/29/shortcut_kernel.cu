#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= size) return;
int i = id % minw;
id /= minw;
int j = id % minh;
id /= minh;
int k = id % minc;
id /= minc;
int b = id % batch;

int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
out[out_index] += add[add_index];
}