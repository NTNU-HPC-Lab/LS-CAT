#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(int *a, int *b, int *c, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < n) {
c[i] = a[i] + b[i];
}
}