#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void BlockPrefix(int *a, int k, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
for(int j = i * k + 1; j < i * k + k && j < n; ++j) {
a[j] += a[j - 1];
}
}