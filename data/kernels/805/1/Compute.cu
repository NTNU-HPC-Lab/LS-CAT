#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void Compute(int *a, int k, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int id = i / k * 2 * k + k + i % k;
if(id < n) {
a[id] += a[id - id % k - 1];
}
}