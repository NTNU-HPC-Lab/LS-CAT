#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createAnaglyph_kernel(uchar4 *out_image, const float *left_image, const float *right_image, int width, int height, int pre_shift) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int x_right = x - pre_shift;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 temp;

if (x < width && y < height) {

temp.x = left_image[__mul24(y, width) + x];

if (x_right > 0 && x_right < width) {
temp.y = right_image[__mul24(y, width) + x_right];
temp.z = temp.y;
} else {
temp.y = 0;
temp.z = 0;
}

temp.w = 255;

out_image[__mul24(y, width) + x] = temp;
}
}