#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertKinectDisparityToRegularDisparity_kernel( float *d_regularDisparity, int d_regularDisparityPitch, const float *d_KinectDisparity, int d_KinectDisparityPitch, int width, int height) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < width) & (y < height)) { // are we in the image?

float d_in =
*((float *)((char *)d_KinectDisparity + y * d_KinectDisparityPitch) +
x);

float d_out = (d_in == 0.0f) ? nanf("") : -d_in;

*((float *)((char *)d_regularDisparity + y *d_regularDisparityPitch) + x) =
d_out;
}
}