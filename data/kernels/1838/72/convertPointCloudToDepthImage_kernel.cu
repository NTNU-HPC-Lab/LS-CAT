#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPointCloudToDepthImage_kernel( unsigned int *depth_image, const float4 *point_cloud, int n_cols, int n_rows, int n_points, float nodal_point_x, float nodal_point_y, float focal_length_x, float focal_length_y, const float *T, const float *R) {

const int ind = blockIdx.x * blockDim.x + threadIdx.x;

if (ind < n_points) {

// fetch point
float4 point = point_cloud[ind];

// transform to camera frame
float x = R[0] * point.x + R[1] * point.y + R[2] * point.z + T[0];
float y = R[3] * point.x + R[4] * point.y + R[5] * point.z + T[1];
float z = R[6] * point.x + R[7] * point.y + R[8] * point.z + T[2];

float inv_z = 1.0f / z;

// project in image
int x_pix = __float2int_rn(focal_length_x * x * inv_z + nodal_point_x);
int y_pix = __float2int_rn(focal_length_y * y * inv_z + nodal_point_y);

// check if inside image
bool valid =
((x_pix >= 0) && (x_pix < n_cols) && (y_pix >= 0) && (y_pix < n_rows));

if (valid) {
int ind_out = y_pix * n_cols + x_pix;
//      depth_image[ind_out] = (unsigned int)(point.z * 1000.0f);
atomicMin(depth_image + ind_out, (unsigned int)(point.z * 1000.0f));
}
}
}