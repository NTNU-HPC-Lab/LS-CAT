#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static float flow_absolute_residual(float x, float y, float ux, float uy, float d, float fx, float fy, float T0, float T1, float T2, float R0, float R1, float R2) {
float rx = -ux + fx * R1 - y * R2 + ((x * x) * R1) / fx + d * fx * T0 -
d * x * T2 - (x * y * R0) / fx;
float ry = -uy - fy * R0 + x * R2 - d * y * T2 - ((y * y) * R0) / fy +
d * fy * T1 + (x * y * R1) / fy;

return sqrtf(rx * rx + ry * ry);
}
__global__ void normal_eqs_flow_weighted_GPU( float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const float *d_abs_res_scales, float w_flow, float w_ar_flow, const float *d_dTR) {

int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be
// multiple of blocksize

int n_flow = d_n_values_flow[blockIdx.y];
int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
int start_ind = d_start_ind_flow[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_flow) { // is this a valid sample?

// fetch flow and Zbuffer from global memory
float2 u = d_flow_compact[in_ind + start_ind];
float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

// compute coordinates
int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = x - ox;
y = y - oy;

// determine M-estimation weight
float w_rel = is_ar_flow ? w_ar_flow : w_flow;
int s6 = blockIdx.y * 6;
float w = w_rel * flow_absolute_residual(x, y, u.x, u.y, disp, fx, fy,
d_dTR[s6], d_dTR[s6 + 1],
d_dTR[s6 + 2], d_dTR[s6 + 3],
d_dTR[s6 + 4], d_dTR[s6 + 5]);
w /= d_abs_res_scales[blockIdx.y];
w = (w > 1) ? 0 : (1.0f - 2.0f * w * w + w * w * w * w);

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix

A0 += w * (disp * disp * fx * fx);
A1 += w * (-disp * disp * x * fx);
A2 += w * (-disp * x * y);
A3 += w * (disp * fx * fx + disp * x * x);
A4 += w * (-disp * y * fx);
A5 += w * (-disp * disp * y * fy);
A6 += w * (-disp * fy * fy - disp * y * y); //!!!!
A7 += w * (disp * x * fy);
A8 += w * (disp * disp * x * x + disp * disp * y * y);
A9 += w * (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy);
A10 +=
w * (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy);
A11 += w * (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y +
y * y * y * y / (fy * fy));
A12 += w * (-2.0f * x * y - x * x * x * y / (fx * fx) -
x * y * y * y / (fy * fy));
A13 += w * (x * y * y / fx - x * fy - x * y * y / fy);
A14 += w * (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) +
x * x * y * y / (fy * fy));
A15 += w * (-y * fx - x * x * y / fx + x * x * y / fy);
A16 += w * (x * x + y * y);

// B-vector

A17 += w * (disp * u.x * fx);
A18 += w * (disp * u.y * fy);
A19 += w * (-disp * x * u.x - disp * y * u.y);
A20 += w * (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy);
A21 += w * (u.x * fx + x * x * u.x / fx + x * y * u.y / fy);
A22 += w * (-y * u.x + x * u.y);
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

d_CO[out_ind] = A0;
d_CO[out_ind + n_val_accum] = A1;
d_CO[out_ind + 2 * n_val_accum] = A2;
d_CO[out_ind + 3 * n_val_accum] = A3;
d_CO[out_ind + 4 * n_val_accum] = A4;
d_CO[out_ind + 5 * n_val_accum] = A5;
d_CO[out_ind + 6 * n_val_accum] = A6;
d_CO[out_ind + 7 * n_val_accum] = A7;
d_CO[out_ind + 8 * n_val_accum] = A8;
d_CO[out_ind + 9 * n_val_accum] = A9;
d_CO[out_ind + 10 * n_val_accum] = A10;
d_CO[out_ind + 11 * n_val_accum] = A11;
d_CO[out_ind + 12 * n_val_accum] = A12;
d_CO[out_ind + 13 * n_val_accum] = A13;
d_CO[out_ind + 14 * n_val_accum] = A14;
d_CO[out_ind + 15 * n_val_accum] = A15;
d_CO[out_ind + 16 * n_val_accum] = A16;
d_CO[out_ind + 17 * n_val_accum] = A17;
d_CO[out_ind + 18 * n_val_accum] = A18;
d_CO[out_ind + 19 * n_val_accum] = A19;
d_CO[out_ind + 20 * n_val_accum] = A20;
d_CO[out_ind + 21 * n_val_accum] = A21;
d_CO[out_ind + 22 * n_val_accum] = A22;
}