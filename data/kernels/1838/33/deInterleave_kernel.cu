#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deInterleave_kernel(float *d_X_out, float *d_Y_out, float2 *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < width) & (y < height)) { // are we in the image?
float2 XY = *((float2 *)((char *)d_XY_in + y * pitch_in) + x);
*((float *)((char *)d_X_out + y *pitch_out) + x) = XY.x;
*((float *)((char *)d_Y_out + y *pitch_out) + x) = XY.y;
}
}