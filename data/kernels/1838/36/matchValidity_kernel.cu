#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matchValidity_kernel(float *d_flow, float *d_disparity, int n_cols, int n_rows) {
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < n_cols) & (y < n_rows)) // are we in the image?
{
unsigned int ind = x + y * n_cols;
bool valid = (isfinite(d_flow[ind]) && isfinite(d_disparity[ind]));
if (!valid) {
d_flow[ind] = nanf("");
d_flow[ind + n_cols * n_rows] = nanf("");
d_disparity[ind] = nanf("");
}
}
}