#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_disparity_GPU(float *d_CD, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, float w_disp) {

int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be
// multiple of blocksize

int n_disparity = d_n_values_disparity[blockIdx.y];
int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
int start_ind = d_start_ind_disparity[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f, A23 = 0.0f,
A24 = 0.0f, A25 = 0.0f, A26 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_disparity) { // is this a valid sample?

// fetch disparity, Zbuffer and normal from global memory
float disp = d_disparity_compact[in_ind + start_ind];
float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
float Zbuffer = tmp.x;
float nx = tmp.y;
float ny = tmp.z;
float nz = tmp.w;

// compute coordinates
int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind];

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = __fdividef((x - ox), fx);
y = __fdividef((y - oy), fy);

// reconstruct 3D point from disparity

float Zd = -(fx * b) / disp; // arbitrary conversion for now using fx
float Xd = x * Zd;
float Yd = y * Zd;

// reconstruct 3D point from model

float Zm = Zbuffer;
float Xm = x * Zm;
float Ym = y * Zm;

// weight the constraint according to (fx*b)/(Zm*Zm) to convert
// from distance- (mm) to image-units (pixel)
float w2 = fx * b / (Zm * Zm);
w2 *= w2;

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix

A0 += w2 * (nx * nx);
A1 += w2 * (nx * ny);
A2 += w2 * (nx * nz);
A3 += w2 * (Ym * nx * nz - Zm * nx * ny);
A4 += w2 * (Zm * (nx * nx) - Xm * nx * nz);
A5 += w2 * (-Ym * (nx * nx) + Xm * nx * ny);

A6 += w2 * (ny * ny);
A7 += w2 * (ny * nz);
A8 += w2 * (-Zm * (ny * ny) + Ym * ny * nz);
A9 += w2 * (-Xm * ny * nz + Zm * nx * ny);
A10 += w2 * (Xm * (ny * ny) - Ym * nx * ny);

A11 += w2 * (nz * nz);
A12 += w2 * (Ym * (nz * nz) - Zm * ny * nz);
A13 += w2 * (-Xm * (nz * nz) + Zm * nx * nz);
A14 += w2 * (Xm * ny * nz - Ym * nx * nz);

A15 += w2 * ((Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) -
Ym * Zm * ny * nz * 2.0f);
A16 += w2 * (-Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny +
Xm * Zm * ny * nz + Ym * Zm * nx * nz);
A17 += w2 * (-Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz +
Xm * Ym * ny * nz + Ym * Zm * nx * ny);

A18 += w2 * ((Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) -
Xm * Zm * nx * nz * 2.0f);
A19 += w2 * (-Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz +
Xm * Ym * nx * nz + Xm * Zm * nx * ny);

A20 += w2 * ((Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) -
Xm * Ym * nx * ny * 2.0f);

// B-vector

A21 += w2 * (Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny -
Ym * nx * ny + Zd * nx * nz - Zm * nx * nz);
A22 += w2 * (Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny -
Xm * nx * ny + Zd * ny * nz - Zm * ny * nz);
A23 += w2 * (Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz -
Xm * nx * nz + Yd * ny * nz - Ym * ny * nz);
A24 += w2 *
(-Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) -
Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny +
Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz);
A25 += w2 *
(Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) +
Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz +
Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz);
A26 += w2 *
(-Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) -
Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz -
Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz);
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

w_disp *= w_disp; // weight relative to flow

d_CD[out_ind] = w_disp * A0;
d_CD[out_ind + n_val_accum] = w_disp * A1;
d_CD[out_ind + 2 * n_val_accum] = w_disp * A2;
d_CD[out_ind + 3 * n_val_accum] = w_disp * A3;
d_CD[out_ind + 4 * n_val_accum] = w_disp * A4;
d_CD[out_ind + 5 * n_val_accum] = w_disp * A5;
d_CD[out_ind + 6 * n_val_accum] = w_disp * A6;
d_CD[out_ind + 7 * n_val_accum] = w_disp * A7;
d_CD[out_ind + 8 * n_val_accum] = w_disp * A8;
d_CD[out_ind + 9 * n_val_accum] = w_disp * A9;
d_CD[out_ind + 10 * n_val_accum] = w_disp * A10;
d_CD[out_ind + 11 * n_val_accum] = w_disp * A11;
d_CD[out_ind + 12 * n_val_accum] = w_disp * A12;
d_CD[out_ind + 13 * n_val_accum] = w_disp * A13;
d_CD[out_ind + 14 * n_val_accum] = w_disp * A14;
d_CD[out_ind + 15 * n_val_accum] = w_disp * A15;
d_CD[out_ind + 16 * n_val_accum] = w_disp * A16;
d_CD[out_ind + 17 * n_val_accum] = w_disp * A17;
d_CD[out_ind + 18 * n_val_accum] = w_disp * A18;
d_CD[out_ind + 19 * n_val_accum] = w_disp * A19;
d_CD[out_ind + 20 * n_val_accum] = w_disp * A20;
d_CD[out_ind + 21 * n_val_accum] = w_disp * A21;
d_CD[out_ind + 22 * n_val_accum] = w_disp * A22;
d_CD[out_ind + 23 * n_val_accum] = w_disp * A23;
d_CD[out_ind + 24 * n_val_accum] = w_disp * A24;
d_CD[out_ind + 25 * n_val_accum] = w_disp * A25;
d_CD[out_ind + 26 * n_val_accum] = w_disp * A26;
}