#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_flow_GPU(float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow) {

int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be
// multiple of blocksize

int n_flow = d_n_values_flow[blockIdx.y];
int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
int start_ind = d_start_ind_flow[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_flow) { // is this a valid sample?

// fetch flow and Zbuffer from global memory
float2 u = d_flow_compact[in_ind + start_ind];
float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

// compute coordinates
int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = x - ox;
y = y - oy;

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix
A0 += (disp * disp * fx * fx);
A1 += (-disp * disp * x * fx);
A2 += (-disp * x * y);
A3 += (disp * fx * fx + disp * x * x);
A4 += (-disp * y * fx);
A5 += (-disp * disp * y * fy);
A6 += (-disp * fy * fy - disp * y * y); //!!!!
A7 += (disp * x * fy);
A8 += (disp * disp * x * x + disp * disp * y * y);
A9 += (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy);
A10 += (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy);
A11 += (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y +
y * y * y * y / (fy * fy));
A12 += (-2.0f * x * y - x * x * x * y / (fx * fx) -
x * y * y * y / (fy * fy));
A13 += (x * y * y / fx - x * fy - x * y * y / fy);
A14 += (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) +
x * x * y * y / (fy * fy));
A15 += (-y * fx - x * x * y / fx + x * x * y / fy);
A16 += (x * x + y * y);

// B-vector

A17 += (disp * u.x * fx);
A18 += (disp * u.y * fy);
A19 += (-disp * x * u.x - disp * y * u.y);
A20 += (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy);
A21 += (u.x * fx + x * x * u.x / fx + x * y * u.y / fy);
A22 += (-y * u.x + x * u.y);
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

d_CO[out_ind] = A0;
d_CO[out_ind + n_val_accum] = A1;
d_CO[out_ind + 2 * n_val_accum] = A2;
d_CO[out_ind + 3 * n_val_accum] = A3;
d_CO[out_ind + 4 * n_val_accum] = A4;
d_CO[out_ind + 5 * n_val_accum] = A5;
d_CO[out_ind + 6 * n_val_accum] = A6;
d_CO[out_ind + 7 * n_val_accum] = A7;
d_CO[out_ind + 8 * n_val_accum] = A8;
d_CO[out_ind + 9 * n_val_accum] = A9;
d_CO[out_ind + 10 * n_val_accum] = A10;
d_CO[out_ind + 11 * n_val_accum] = A11;
d_CO[out_ind + 12 * n_val_accum] = A12;
d_CO[out_ind + 13 * n_val_accum] = A13;
d_CO[out_ind + 14 * n_val_accum] = A14;
d_CO[out_ind + 15 * n_val_accum] = A15;
d_CO[out_ind + 16 * n_val_accum] = A16;
d_CO[out_ind + 17 * n_val_accum] = A17;
d_CO[out_ind + 18 * n_val_accum] = A18;
d_CO[out_ind + 19 * n_val_accum] = A19;
d_CO[out_ind + 20 * n_val_accum] = A20;
d_CO[out_ind + 21 * n_val_accum] = A21;
d_CO[out_ind + 22 * n_val_accum] = A22;
}