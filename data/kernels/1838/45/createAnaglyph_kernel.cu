#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static float rgbaToGray(uchar4 rgba) {
return (0.299f * (float)rgba.x + 0.587f * (float)rgba.y +
0.114f * (float)rgba.z);
}
__global__ void createAnaglyph_kernel(uchar4 *out_image, const uchar4 *left_image, const uchar4 *right_image, int width, int height, int pre_shift) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int x_right = x - pre_shift;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
uchar4 temp;

if (x < width && y < height) {

temp.x = rgbaToGray(left_image[y * width + x]);

if (x_right > 0 && x_right < width) {
temp.y = rgbaToGray(right_image[y * width + x_right]);
temp.z = temp.y;
} else {
temp.y = 0;
temp.z = 0;
}

temp.w = 255;

out_image[y * width + x] = temp;
}
}