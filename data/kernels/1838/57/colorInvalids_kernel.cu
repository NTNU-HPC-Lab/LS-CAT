#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorInvalids_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

if (x < width && y < height) {
int ind = __mul24(y, width) + x;
uchar4 temp = out_image[ind];
float value = in_image[ind];

if (!isfinite(value)) { // color
temp.x *= 0.5f;
temp.y *= 0.5f;
}

out_image[ind] = temp;
}
}