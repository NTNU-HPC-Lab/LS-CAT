#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_normal_eqs_64_mult_constr_GPU(float *d_C_reduced, const float *d_C, int gridDim_x_normal_equations, int n_constraints) {
// check if there are constraints left to be processed
int constraint_ind = blockIdx.x * 4 + threadIdx.y;

if (constraint_ind < n_constraints) {

int tid = 64 * threadIdx.y + threadIdx.x;

// put data in shared memory
int ind = blockIdx.y * n_constraints * gridDim_x_normal_equations * 64 +
constraint_ind * gridDim_x_normal_equations * 64 + threadIdx.x;

__shared__ float DATA[64 * 4];

// load and sum the first gridDim_x_normal_equations elements
float tmp = 0.0f;
for (int i = 0; i < gridDim_x_normal_equations; i++)
tmp += d_C[ind + i * 64];
DATA[tid] = tmp;

__syncthreads(); // ensure reading stage has finished

if ((tid - 64 * threadIdx.y) < 32) { // warp-reduce
DATA[tid] += DATA[tid + 32];
__syncthreads();
DATA[tid] += DATA[tid + 16];
__syncthreads();
DATA[tid] += DATA[tid + 8];
__syncthreads();
DATA[tid] += DATA[tid + 4];
__syncthreads();
DATA[tid] += DATA[tid + 2];
__syncthreads();
DATA[tid] += DATA[tid + 1];
__syncthreads();
}

// write results
if (threadIdx.x == 0)
d_C_reduced[blockIdx.y * n_constraints + constraint_ind] = DATA[tid];
}
}