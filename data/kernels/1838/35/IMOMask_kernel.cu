#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IMOMask_kernel(float *d_IMOMask, float *d_IMO, const float *d_disparity, float offset, int n_cols, int n_rows) {
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < n_cols) & (y < n_rows)) // are we in the image?
{
unsigned int ind = x + y * n_cols;
if (!(bool)(d_IMOMask[ind])) {
d_IMO[ind] = nanf("");
}
}
}