#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertKinectDisparityInPlace_kernel(float *d_disparity, int pitch, int width, int height, float depth_scale) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < width) & (y < height)) { // are we in the image?

float *d_in = (float *)((char *)d_disparity + y * pitch) + x;
*d_in = (*d_in == 0.0f) ? nanf("") : (-depth_scale / *d_in);
}
}