#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
uchar4 temp;
if (x < width && y < height) {
int IND = y * width + x;
float val = in_image[IND];
temp.x = val;
temp.y = val;
temp.z = val;
temp.w = 255;
out_image[IND] = temp;
}
}