#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_normal_eqs_64_GPU(float *d_C_reduced, float *d_C, int gridDim_x_normal_equations) {

int tid = threadIdx.x;
int bx = blockIdx.x;
// put data in shared memory

int ind = blockIdx.y * gridDim.x * gridDim_x_normal_equations * 64 +
bx * gridDim_x_normal_equations * 64 + tid;

__shared__ float DATA[64];

// load and sum the first 20 elements
float tmp = 0.0f;
for (int i = 0; i < gridDim_x_normal_equations; i++)
tmp += d_C[ind + i * 64];
DATA[tid] = tmp;

__syncthreads(); // ensure reading stage has finished

// reduction
if (tid < 32) { // warp-reduce
DATA[tid] += DATA[tid + 32];
__syncthreads();
DATA[tid] += DATA[tid + 16];
__syncthreads();
DATA[tid] += DATA[tid + 8];
__syncthreads();
DATA[tid] += DATA[tid + 4];
__syncthreads();
DATA[tid] += DATA[tid + 2];
__syncthreads();
DATA[tid] += DATA[tid + 1];
__syncthreads();
}

// write results
if (tid == 0)
d_C_reduced[blockIdx.y * gridDim.x + bx] = DATA[0];
}