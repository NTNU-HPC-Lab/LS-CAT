#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPitchedFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 temp;
if (x < width && y < height) {
float val = *((float *)((char *)in_image + y * pitch) + x);

// first draw unmatched pixels in white
if (!isfinite(val)) {
temp.x = 255;
temp.y = 255;
temp.z = 255;
temp.w = 255;
} else {
// rescale value from [lowerLim,upperLim] to [0,1]
val -= lowerLim;
val /= (upperLim - lowerLim);

float r = 1.0f;
float g = 1.0f;
float b = 1.0f;
if (val < 0.25f) {
r = 0;
g = 4.0f * val;
} else if (val < 0.5f) {
r = 0;
b = 1.0 + 4.0f * (0.25f - val);
} else if (val < 0.75f) {
r = 4.0f * (val - 0.5f);
b = 0;
} else {
g = 1.0f + 4.0f * (0.75f - val);
b = 0;
}
temp.x = 255.0 * r;
temp.y = 255.0 * g;
temp.z = 255.0 * b;
temp.w = 255;
}
out_image[__mul24(y, width) + x] = temp;
}
}