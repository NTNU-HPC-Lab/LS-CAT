#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiple_median_reduce_shuffle_gpu(const float *d_in, float *d_out, const float *d_random_numbers, const int *d_start_inds, const int *d_n_in) {

/**************/
/* initialize */
/**************/

int segment = blockIdx.y;

// compute indices

int t_ind = threadIdx.x;
int g_ind =
blockIdx.x * MED_BLOCK_SIZE +
t_ind; // means that every row of blocks uses the same random numbers

// allocate shared memory

//  __shared__ float DATA[MED_BLOCK_SIZE];
__shared__ float DATA[256];

/**************/
/* load stage */
/**************/

if (t_ind < MED_BLOCK_SIZE) {
int sample_ind = d_start_inds[segment] +
floorf(d_random_numbers[g_ind] * (float)d_n_in[segment]);
DATA[t_ind] = d_in[sample_ind];
}

__syncthreads();

/*******************/
/* reduction stage */
/*******************/

for (int s = 1; s < MED_BLOCK_SIZE; s *= 3) {

int index = 3 * s * t_ind;

if (index < MED_BLOCK_SIZE) {

// fetch three values
float value1 = DATA[index];
float value2 = DATA[index + s];
float value3 = DATA[index + 2 * s];

// extract the middle value (median)
float smallest = fminf(value1, value2);
value2 = fmaxf(value1, value2);
value1 = smallest;

value3 = fmaxf(value1, value3);
value2 = fminf(value2, value3);

DATA[index] = value2;
}

__syncthreads();
}

/***************/
/* write stage */
/***************/

// write this block's approx median (first element)

if (t_ind == 0) {
d_out[gridDim.x * blockIdx.y + blockIdx.x] = DATA[0];
}
}