#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
int x_kernel[3][3] = { { 1, 0, -1 }, { 2, 0, -2 }, { 1, 0, -1 } };

int index = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = index * chunk_size_per_thread; i < (index + 1) * chunk_size_per_thread - 1; i++) {
if (i + 2 * image_width + 1 < image_width * image_height) {
if (i == 0 && blockIdx.x == 0 && threadIdx.x == 0) {
continue;
} else {
x_gradient[i] =
x_kernel[0][0] * channel[i - 1] +
x_kernel[1][0] * channel[i] +
x_kernel[2][0] * channel[i + 1] +
x_kernel[0][1] * channel[i + image_width - 1] +
x_kernel[1][1] * channel[i + image_width] +
x_kernel[2][1] * channel[i + image_width + 1] +
x_kernel[0][2] * channel[i + 2 * image_width - 1] +
x_kernel[1][2] * channel[i + 2 * image_width] +
x_kernel[2][2] * channel[i + 2 * image_width + 1];
}
}
}
return;
}