#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;




__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height) {
int x_kernel[3][3] = { { 1, 0, -1 }, { 2, 0, -2 }, { 1, 0, -1 } };
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index == 0) {
return;
}
x_gradient[index] =
x_kernel[0][0] * channel[index - 1] +
x_kernel[1][0] * channel[index] +
x_kernel[2][0] * channel[index + 1] +
x_kernel[0][1] * channel[index + image_width - 1] +
x_kernel[1][1] * channel[index + image_width] +
x_kernel[2][1] * channel[index + image_width + 1] +
x_kernel[0][2] * channel[index + 2 * image_width - 1] +
x_kernel[1][2] * channel[index + 2 * image_width] +
x_kernel[2][2] * channel[index + 2 * image_width + 1];
return;
}