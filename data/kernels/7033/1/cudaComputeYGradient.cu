#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;




__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height) {
int y_kernel[3][3] = { { 1, 2, 1 }, { 0, 0, 0 }, { -1, -2, -1 } };
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index == 0) {
return;
}
y_gradient[index] =
y_kernel[0][0] * channel[index - 1] +
y_kernel[1][0] * channel[index] +
y_kernel[2][0] * channel[index + 1] +
y_kernel[0][1] * channel[index + image_width - 1] +
y_kernel[1][1] * channel[index + image_width] +
y_kernel[2][1] * channel[index + image_width + 1] +
y_kernel[0][2] * channel[index + 2 * image_width - 1] +
y_kernel[1][2] * channel[index + 2 * image_width] +
y_kernel[2][2] * channel[index + 2 * image_width + 1];
return;
}