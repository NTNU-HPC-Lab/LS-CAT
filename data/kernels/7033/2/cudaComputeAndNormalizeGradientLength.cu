#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;




__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int gradient_length = int(sqrt(float(x_gradient[index] * x_gradient[index] + y_gradient[index] * y_gradient[index])));
if (gradient_length > 255) {
gradient_length = 255;
}
channel_values[index] = gradient_length;
return;
}