#include "hip/hip_runtime.h"
#include "includes.h"
/*
Detected 1 CUDA Capable device(s)

Device 0: "GeForce GT 320M"
CUDA Driver Version / Runtime Version          5.0 / 5.0
CUDA Capability Major/Minor version number:    1.2
Total amount of global memory:                 1024 MBytes (1073741824 bytes)
( 3) Multiprocessors x (  8) CUDA Cores/MP:    24 CUDA Cores
GPU Clock rate:                                1100 MHz (1.10 GHz)
Memory Clock rate:                             790 Mhz
Memory Bus Width:                              128-bit
Max Texture Dimension Size (x,y,z)             1D=(8192), 2D=(65536,32768), 3D=(2048,2048,2048)
Max Layered Texture Size (dim) x layers        1D=(8192) x 512, 2D=(8192,8192) x 512
Total amount of constant memory:               65536 bytes
Total amount of shared memory per block:       16384 bytes
Total number of registers available per block: 16384
Warp size:                                     32
Maximum number of threads per multiprocessor:  1024
Maximum number of threads per block:           512
Maximum sizes of each dimension of a block:    512 x 512 x 64
Maximum sizes of each dimension of a grid:     65535 x 65535 x 1
Maximum memory pitch:                          2147483647 bytes
Texture alignment:                             256 bytes
Concurrent copy and kernel execution:          Yes with 1 copy engine(s)
Run time limit on kernels:                     Yes
Integrated GPU sharing Host Memory:            No
Support host page-locked memory mapping:       Yes
Alignment requirement for Surfaces:            Yes
Device has ECC support:                        Disabled
CUDA Device Driver Mode (TCC or WDDM):         WDDM (Windows Display Driver Model)
Device supports Unified Addressing (UVA):      No
Device PCI Bus ID / PCI location ID:           2 / 0
Compute Mode:
< Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >

deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 5.0, CUDA Runtime Version = 5.0, NumDevs = 1, Device0 = GeForce GT 320M
*/




__global__ void freqencyMethod2(char *d_dat,int len, int *d_freq)
{//·½·¨¶þ£¬ÏÈ½«Êý¾ÝÔ­×Ó¼Óµ½share memoryÖÐ£¬È»ºóÔÙÀÛ¼Óµ½ÏÔ´æÉÏ¡£

__shared__ int sfreq[26];//

if(threadIdx.x < 26)
sfreq[threadIdx.x] = 0;////ÏÈÇå¿Õ¡£
__syncthreads();
int gridsize = blockDim.x * gridDim.x;
int pos = 0;
for(int i=threadIdx.x + blockIdx.x*blockDim.x; i< len; i += gridsize)
{
pos = d_dat[i]-'a';
atomicAdd(&sfreq[pos],1);
}
__syncthreads();

if(threadIdx.x<26)///Èç¹ûÏÔ¿¨Ö§³ÖÔ­×Ó¼Ó£¬¿ÉÒÔÊ¹ÓÃÔ­×Ó¼Ó£¬Ö±½Ó¼Óµ½ÏÔ´æÉÏ¡£ÄÇÑù¾ÍÃ»ÓÐµÚ¶þ²½¡£ 1.1¼°ÒÔÉÏÖ§³ÖÈ«¾ÖÏÔ´æµÄ32Î»Ô­×Ó²Ù×÷¡£
atomicAdd(&d_freq[threadIdx.x],sfreq[threadIdx.x]);

}