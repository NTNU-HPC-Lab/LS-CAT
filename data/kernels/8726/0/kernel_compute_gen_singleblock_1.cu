#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__device__ void gpu_swap(void **a, void **b) {
void *temp = *a;
*a = *b;
*b = temp;
}
__global__ void kernel_compute_gen_singleblock_1(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols, uint32_t iterations) {
uint32_t cell = threadIdx.x;

uint32_t x1 = cell     & cols-1; //% cols;
uint32_t x0 = (x1 - 1) & cols-1; //% cols;
uint32_t x2 = (x1 + 1) & cols-1; //% cols;

uint32_t y1 = cell - x1;
uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

for (uint32_t iter = 0; iter < iterations; iter++){
uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] + matrix_src[x0 + y1] +
matrix_src[x2 + y1] + matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];
matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0;
gpu_swap((void**)&matrix_dst,(void**)&matrix_src);
__syncthreads();
}
}