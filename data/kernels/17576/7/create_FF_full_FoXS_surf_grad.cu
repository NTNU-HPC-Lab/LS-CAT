#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void create_FF_full_FoXS_surf_grad ( float *FF_table, float *V, float c2, int *Ele, float *FF_full, float *surf_grad, int num_q, int num_ele, int num_atom, int num_atom2) {

__shared__ float FF_pt[7];
float hydration;
for (int ii = blockIdx.x; ii < num_q+1; ii += gridDim.x) {

// Get form factor for this block (or q vector)
if (ii < num_q) {
for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
}
}
__syncthreads();

// In FoXS since c2 remains the same for all elements it is reduced to one value.
hydration = c2 * FF_pt[num_ele];
//if (ii == num_q && threadIdx.x == 0) {printf("Hydration is: %6.3f\n", hydration);}
__syncthreads();
// Calculate atomic form factor for this q
// However to keep compatible to HyPred method we leave atom type def unchanged.
if (ii == num_q) {
// calculate surf_grad
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
//int atomt = Ele[jj];
//printf("B surf grads = %6.3f, %6.3f, %6.3f. \n",
//       surf_grad[3*jj], surf_grad[3*jj+1], surf_grad[3*jj+2]);
/*surf_grad[3*jj]   *= hydration;
surf_grad[3*jj+1] *= hydration;
surf_grad[3*jj+2] *= hydration;*/
surf_grad[3*jj]   *= c2;
surf_grad[3*jj+1] *= c2;
surf_grad[3*jj+2] *= c2;
//printf("A surf grads = %6.3f, %6.3f, %6.3f. \n",
//       surf_grad[3*jj], surf_grad[3*jj+1], surf_grad[3*jj+2]);
}
} else {
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
int atomt = Ele[jj];
if (atomt > 5) {  // Which means this is a hydrogen
FF_full[ii*num_atom2 + jj] = FF_pt[0];
FF_full[ii*num_atom2 + jj] += hydration * V[jj];
} else {          // Heavy atoms - do the same as before
FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
FF_full[ii*num_atom2 + jj] += hydration * V[jj];
}
}
}
if (threadIdx.x == 0) FF_full[ii * num_atom2 + num_atom + 1] = FF_pt[num_ele];
}
}