#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void force_calc ( float *Force, int num_atom, int num_q, float *f_ptxc, float *f_ptyc, float *f_ptzc, int num_atom2, int num_q2, int *Ele, float force_ramp) {
// Do column tree sum of f_ptxc for f_ptx for every atom, then assign threadIdx.x == 0 (3 * num_atoms) to Force. Force is num_atom * 3.
if (blockIdx.x >= num_atom) return;
for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
__syncthreads();
for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
f_ptxc[ii + iAccum * num_atom2] += f_ptxc[ii + iAccum * num_atom2 + stride * num_atom2];
f_ptyc[ii + iAccum * num_atom2] += f_ptyc[ii + iAccum * num_atom2 + stride * num_atom2];
f_ptzc[ii + iAccum * num_atom2] += f_ptzc[ii + iAccum * num_atom2 + stride * num_atom2];
}
}
__syncthreads();
if (threadIdx.x == 0) {
if (Ele[ii]) {
Force[ii*3    ] = f_ptxc[ii] * force_ramp;
Force[ii*3 + 1] = f_ptyc[ii] * force_ramp;
Force[ii*3 + 2] = f_ptzc[ii] * force_ramp;
}
}
__syncthreads();
}
}