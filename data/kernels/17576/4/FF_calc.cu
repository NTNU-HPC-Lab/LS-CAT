#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FF_calc ( float *q_S_ref_dS, float *WK, float *vdW, int num_q, int num_ele, float c1, float r_m, float *FF_table, float rho) {

__shared__ float q_pt, q_WK, C1, expC1;
__shared__ float FF_pt[7]; // num_ele + 1, the last one for water.
__shared__ float vdW_s[7];
__shared__ float WK_s[66];
__shared__ float C1_PI_43_rho;
if (blockIdx.x >= num_q) return; // out of q range
for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
q_pt = q_S_ref_dS[ii];
q_WK = q_pt / 4.0 / PI;
// FoXS C1 term
expC1 = -powf(4.0 * PI / 3.0, 1.5) * q_WK * q_WK * r_m * r_m * (c1 * c1 - 1.0) / 4.0 / PI;
C1 = powf(c1,3) * exp(expC1);
C1_PI_43_rho = C1 * PI * 4.0 / 3.0 * rho;
for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
WK_s[jj] = WK[jj];
}
__syncthreads();

// Calculate Form factor for this block (or q vector)
for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
vdW_s[jj] = vdW[jj];
if (jj == num_ele) {
// water
FF_pt[jj] = WK_s[3*11+5];
FF_pt[jj] += 2.0 * WK_s[5];
FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
for (int kk = 0; kk < 5; kk ++) {
FF_pt[jj] += WK_s[3*11+kk] * exp(-WK_s[3*11+kk+6] * q_WK * q_WK);
FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK);
FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK);
}
} else {
FF_pt[jj] = WK_s[jj*11+5];
// The part is for excluded volume
FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
for (int kk = 0; kk < 5; kk++) {
FF_pt[jj] += WK_s[jj*11+kk] * exp(-WK_s[jj*11+kk+6] * q_WK * q_WK);
}
}
FF_table[ii*(num_ele+1)+jj] = FF_pt[jj];
}
}
}