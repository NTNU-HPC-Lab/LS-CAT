#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "FF_calc.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *q_S_ref_dS = NULL;
hipMalloc(&q_S_ref_dS, XSIZE*YSIZE);
float *WK = NULL;
hipMalloc(&WK, XSIZE*YSIZE);
float *vdW = NULL;
hipMalloc(&vdW, XSIZE*YSIZE);
int num_q = 1;
int num_ele = 1;
float c1 = 1;
float r_m = 1;
float *FF_table = NULL;
hipMalloc(&FF_table, XSIZE*YSIZE);
float rho = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
FF_calc<<<gridBlock,threadBlock>>>(q_S_ref_dS,WK,vdW,num_q,num_ele,c1,r_m,FF_table,rho);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
FF_calc<<<gridBlock,threadBlock>>>(q_S_ref_dS,WK,vdW,num_q,num_ele,c1,r_m,FF_table,rho);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
FF_calc<<<gridBlock,threadBlock>>>(q_S_ref_dS,WK,vdW,num_q,num_ele,c1,r_m,FF_table,rho);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}