#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_V ( float *V, float *V_s, int num_atom, int num_atom2, int *Ele, float *vdW) {

for (int ii = threadIdx.x; ii < num_atom2; ii += blockDim.x) {
if (ii < num_atom) {
int atomi = Ele[ii];
if (atomi > 5) atomi = 0;
V_s[ii] = V[ii] * 4.0 * PI * vdW[atomi] * vdW[atomi];
} else {
V_s[ii] = 0.0;
}
}
for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
__syncthreads();
for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
V_s[iAccum] += V_s[stride + iAccum];
}
}
__syncthreads();
if (threadIdx.x == 0) printf("Convex contact area = %.3f A^2.\n", V_s[0]);
}