#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void force_calc_EMA ( float *Force, double *Force_old, int num_atom, int num_q, float *f_ptxc, float *f_ptyc, float *f_ptzc, int num_atom2, int num_q2, int *Ele, double EMA_norm, float force_ramp) {
// Do column tree sum of f_ptxc for f_ptx for every atom, then assign threadIdx.x == 0 (3 * num_atoms) to Force. Force is num_atom * 3.
if (blockIdx.x >= num_atom) return;
for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
__syncthreads();
for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
f_ptxc[ii + iAccum * num_atom2] += f_ptxc[ii + iAccum * num_atom2 + stride * num_atom2];
f_ptyc[ii + iAccum * num_atom2] += f_ptyc[ii + iAccum * num_atom2 + stride * num_atom2];
f_ptzc[ii + iAccum * num_atom2] += f_ptzc[ii + iAccum * num_atom2 + stride * num_atom2];
}
}
__syncthreads();
if (threadIdx.x == 0) {
if (Ele[ii]) {
Force_old[ii*3    ] *= (EMA_norm - 1.0);
Force_old[ii*3    ] -= (double)f_ptxc[ii];
Force_old[ii*3    ] /= EMA_norm;
Force_old[ii*3 + 1] *= (EMA_norm - 1.0);
Force_old[ii*3 + 1] -= (double)f_ptyc[ii];
Force_old[ii*3 + 1] /= EMA_norm;
Force_old[ii*3 + 2] *= (EMA_norm - 1.0);
Force_old[ii*3 + 2] -= (double)f_ptzc[ii];
Force_old[ii*3 + 2] /= EMA_norm;
Force[ii*3    ] = (float)Force_old[ii*3    ] * force_ramp;
Force[ii*3 + 1] = (float)Force_old[ii*3 + 1] * force_ramp;
Force[ii*3 + 2] = (float)Force_old[ii*3 + 2] * force_ramp;
}
}
__syncthreads();
}
}