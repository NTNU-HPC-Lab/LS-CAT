#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dist_calc ( float *coord, int *close_num, int *close_flag, int *close_idx, int num_atom, int num_atom2) {

// close_flag is a 1024 x num_atom2 int matrix initialized to 0.
// close_idx: A num_atom x 200 int matrix, row i of which only the first close_num[i] elements are defined. (Otherwise it's -1).
__shared__ float x_ref, y_ref, z_ref;
__shared__ int idz;
__shared__ int temp[2048];
// Calc distance
for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
if (threadIdx.x == 0) {
x_ref = coord[3*ii  ];
y_ref = coord[3*ii+1];
z_ref = coord[3*ii+2];
}
int idy = ii % gridDim.x; // This will be what row of close_flag this block is putting its value in.
__syncthreads();
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
float r2t = (coord[3*jj  ] - x_ref) * (coord[3*jj  ] - x_ref) +
(coord[3*jj+1] - y_ref) * (coord[3*jj+1] - y_ref) +
(coord[3*jj+2] - z_ref) * (coord[3*jj+2] - z_ref);

if (r2t < 34.0) {
close_flag[idy*num_atom2+jj] = 1; // roughly 2 A + 2 A vdW + 2 * 1.8 A probe
} else {
close_flag[idy*num_atom2+jj] = 0;
}
if (ii == jj) close_flag[idy*num_atom2+jj] = 0;
}
__syncthreads();
// Do pre scan
idz = 0;
int temp_sum = 0;
for (int jj = threadIdx.x; jj < num_atom2; jj += 2 * blockDim.x) {
int idx = jj % blockDim.x;
int offset = 1;
temp[2 * idx]     = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx];
temp[2 * idx + 1] = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1];
for (int d = 2 * blockDim.x>>1; d > 0; d >>= 1) { // up-sweep
__syncthreads();
if (idx < d) {
int ai = offset * (2 * idx + 1) - 1;
int bi = offset * (2 * idx + 2) - 1;
temp[bi] += temp[ai];
}
offset *= 2;
}
__syncthreads();
temp_sum = close_num[ii];
__syncthreads();
if (idx == 0) {
close_num[ii] += temp[2 * blockDim.x - 1]; // log the total number of 1's in this blockDim
temp[2 * blockDim.x - 1] = 0;
}
__syncthreads();
for (int d = 1; d < blockDim.x * 2; d *= 2) { //down-sweep
offset >>= 1;
__syncthreads();
if (idx < d) {
int ai = offset * (2 * idx + 1) - 1;
int bi = offset * (2 * idx + 2) - 1;
int t    = temp[ai];
temp[ai] = temp[bi];
temp[bi] += t;
}
}

__syncthreads();

// Finally assign the indices
if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx] == 1) {
close_idx[ii * 1024 + temp[2*idx] + temp_sum] = 2 * idx + 2 * blockDim.x * idz;
}
if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1] == 1) {
close_idx[ii * 1024 + temp[2*idx+1] + temp_sum] = 2*idx+1 + 2 * blockDim.x * idz;
}
idz++;
__syncthreads();
}
}
}