#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void create_FF_full_HyPred ( float *FF_table, float *V, float c2_F, float *c2_H, int *Ele, float *FF_full, int num_q, int num_ele, int num_atom, int num_atom2) {

__shared__ float FF_pt[7];
__shared__ float hydration[10];
for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

// Get form factor for this block (or q vector)
if (ii < num_q) {
for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
}
}
__syncthreads();

for (int jj = threadIdx.x; jj < 10; jj += blockDim.x) {
hydration[jj] = c2_F * c2_H[jj] * FF_pt[num_ele];
}
__syncthreads();

// Calculate atomic form factor for this q
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
int atomt = Ele[jj];
if (atomt > 5) {  // Which means this is a hydrogen
FF_full[ii*num_atom2 + jj] = FF_pt[0];
FF_full[ii*num_atom2 + jj] += hydration[atomt] * V[jj];
} else { // Heavy atoms - do the same as before
FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
FF_full[ii*num_atom2 + jj] += hydration[atomt] * V[jj];
}
}
}
}