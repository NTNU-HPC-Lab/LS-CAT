#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "create_FF_full_FoXS.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *FF_table = NULL;
hipMalloc(&FF_table, XSIZE*YSIZE);
float *V = NULL;
hipMalloc(&V, XSIZE*YSIZE);
float c2 = 1;
int *Ele = NULL;
hipMalloc(&Ele, XSIZE*YSIZE);
float *FF_full = NULL;
hipMalloc(&FF_full, XSIZE*YSIZE);
int num_q = 1;
int num_ele = 1;
int num_atom = 1;
int num_atom2 = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
create_FF_full_FoXS<<<gridBlock,threadBlock>>>(FF_table,V,c2,Ele,FF_full,num_q,num_ele,num_atom,num_atom2);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
create_FF_full_FoXS<<<gridBlock,threadBlock>>>(FF_table,V,c2,Ele,FF_full,num_q,num_ele,num_atom,num_atom2);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
create_FF_full_FoXS<<<gridBlock,threadBlock>>>(FF_table,V,c2,Ele,FF_full,num_q,num_ele,num_atom,num_atom2);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}