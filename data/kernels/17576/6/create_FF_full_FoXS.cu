#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void create_FF_full_FoXS ( float *FF_table, float *V, float c2, int *Ele, float *FF_full, int num_q, int num_ele, int num_atom, int num_atom2) {

__shared__ float FF_pt[7];
float hydration;
for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

// Get form factor for this block (or q vector)
if (ii < num_q) {
for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
}
}
__syncthreads();

// In FoXS since c2 remains the same for all elements it is reduced to one value.
hydration = c2 * FF_pt[num_ele];

// Calculate atomic form factor for this q
// However to keep compatible to HyPred method we leave atom type def unchanged.
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
int atomt = Ele[jj];
if (atomt > 5) {  // Which means this is a hydrogen
FF_full[ii*num_atom2 + jj] = FF_pt[0];
FF_full[ii*num_atom2 + jj] += hydration * V[jj];
} else {          // Heavy atoms - do the same as before
FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
FF_full[ii*num_atom2 + jj] += hydration * V[jj];
}
}
}
}