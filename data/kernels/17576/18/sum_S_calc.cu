#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_S_calc ( float *S_calcc, float *f_ptxc, float *f_ptyc, float *f_ptzc, float *S_calc, float *Aq, float *q_S_ref_dS, int num_q, int num_atom, int num_atom2, float alpha, float k_chi, float *sigma2) {

for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
// Tree-like summation of S_calcc to get S_calc
for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
__syncthreads();
for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
}
}
__syncthreads();

S_calc[ii] = S_calcc[ii * num_atom2];
__syncthreads();
if (threadIdx.x == 0) {
Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
Aq[ii] *= -alpha;
Aq[ii] += q_S_ref_dS[ii + 2*num_q];
Aq[ii] *= k_chi / sigma2[ii];
Aq[ii] += Aq[ii];
}
__syncthreads();
for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
}
}
}