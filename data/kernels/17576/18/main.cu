#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "sum_S_calc.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *S_calcc = NULL;
hipMalloc(&S_calcc, XSIZE*YSIZE);
float *f_ptxc = NULL;
hipMalloc(&f_ptxc, XSIZE*YSIZE);
float *f_ptyc = NULL;
hipMalloc(&f_ptyc, XSIZE*YSIZE);
float *f_ptzc = NULL;
hipMalloc(&f_ptzc, XSIZE*YSIZE);
float *S_calc = NULL;
hipMalloc(&S_calc, XSIZE*YSIZE);
float *Aq = NULL;
hipMalloc(&Aq, XSIZE*YSIZE);
float *q_S_ref_dS = NULL;
hipMalloc(&q_S_ref_dS, XSIZE*YSIZE);
int num_q = 1;
int num_atom = 1;
int num_atom2 = 1;
float alpha = 2;
float k_chi = 1;
float *sigma2 = NULL;
hipMalloc(&sigma2, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sum_S_calc<<<gridBlock,threadBlock>>>(S_calcc,f_ptxc,f_ptyc,f_ptzc,S_calc,Aq,q_S_ref_dS,num_q,num_atom,num_atom2,alpha,k_chi,sigma2);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
sum_S_calc<<<gridBlock,threadBlock>>>(S_calcc,f_ptxc,f_ptyc,f_ptzc,S_calc,Aq,q_S_ref_dS,num_q,num_atom,num_atom2,alpha,k_chi,sigma2);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
sum_S_calc<<<gridBlock,threadBlock>>>(S_calcc,f_ptxc,f_ptyc,f_ptzc,S_calc,Aq,q_S_ref_dS,num_q,num_atom,num_atom2,alpha,k_chi,sigma2);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}