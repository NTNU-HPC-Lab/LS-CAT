#include "hip/hip_runtime.h"
#include "includes.h"



__global__ static void calc_linear_kernel_predict(int objs,int coords,double* x,int objs_train,double* x_train,double* out){
int id=blockDim.x * blockIdx.x + threadIdx.x;
int i=id/objs;
int j=id%objs;
if (i<objs_train){
double r=1.0;
for (int k=0;k<coords;k++){
r += x_train[coords*i+k] * x[coords*j+k];
}
out[id]=r;
}
}