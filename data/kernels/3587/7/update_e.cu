#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void update_e(int objs,double* e,double* kval,double b_old,double b_new,int i,int j,int yi,int yj,double ai_old,double ai_new,double aj_old,double aj_new){
int id=blockDim.x * blockIdx.x + threadIdx.x;
if (id<objs){
double val=e[id];
val+=(b_new-b_old);
double ti=yi*kval[i*objs+id];
double tj=yj*kval[j*objs+id];
val += ti*(ai_new-ai_old);
val += tj*(aj_new-aj_old);
e[id]=val;
}
}