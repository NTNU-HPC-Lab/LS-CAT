#include "hip/hip_runtime.h"
#include "includes.h"



__global__ static void calc_predict(int objs,int objs_train,double* a,double b,int* y_train,double* kval,int* y){
int id=blockDim.x * blockIdx.x + threadIdx.x;
if (id<objs){
double fx=b;
for (int i=0;i<objs_train;i++){
//access to a and y are not coalesced
fx+=a[i]*y_train[i]*kval[i*objs+id];
}
y[id] = fx>=0 ? 1:-1;
}
}