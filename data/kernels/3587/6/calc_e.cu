#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void calc_e(int objs,double* a,double b,int* y,double* kval,double* e){
int id=blockDim.x * blockIdx.x + threadIdx.x;
if (id<objs){
double fx=b;
for (int i=0;i<objs;i++){
//access to a and y are not coalesced
fx+=a[i]*y[i]*kval[i*objs+id];
}
e[id]=fx-y[id];
}
}