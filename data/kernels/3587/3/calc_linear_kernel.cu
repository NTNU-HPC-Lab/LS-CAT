#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void calc_linear_kernel(int objs,int coords,double* x,double* out){
int id=blockDim.x * blockIdx.x + threadIdx.x;
int i=id/objs;
int j=id%objs;
if (i<objs){

double r=0.0;
for (int k=0;k<coords;k++){
r+=x[objs*k+i]*x[objs*k+j];
}
out[objs*i+j]=r;
}
}