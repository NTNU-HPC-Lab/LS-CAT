#include "hip/hip_runtime.h"
#include "includes.h"

//#define NDEBUG


const static float eps = 1e-6;
const static size_t blocSize = 8;
const static size_t size = 1024;



__global__ void matMultiply1D(float* matA, float* matB, float* Dest, int dimensions)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
if (i < dimensions)
{
float vectA[2048];
for (unsigned k = 0; k != dimensions; ++k)
{
vectA[k] = matB[i*dimensions + k];
}
for (unsigned j = 0; j != dimensions; ++j)
{
float res = 0.0f;
for (unsigned k = 0; k != dimensions; ++k)
{
res += vectA[k] * matB[k*dimensions + j];
}
Dest[i*dimensions + j] = res;
}
}
}