#include "hip/hip_runtime.h"
#include "includes.h"

//#define NDEBUG


const static float eps = 1e-6;
const static size_t blocSize = 8;
const static size_t size = 1024;



__global__ void matMultiply2D(float* matA, float* matB, float* Dest, int dimensions)
{
int ix = threadIdx.x + blockIdx.x*blockDim.x;
int iy = threadIdx.y + blockIdx.y*blockDim.y;

if (ix < dimensions&&iy < dimensions)
{
float res = 0.0f;
for (unsigned k = 0; k != dimensions; ++k)
{
res += matA[ix*dimensions + k] * matB[k*dimensions + iy];
}
Dest[ix*dimensions + iy] = res;
}
}