#include "hip/hip_runtime.h"
#include "includes.h"



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);





__global__ void global_scan(float* d_out, float* d_in)
{
int index = threadIdx.x;
float out = 0.00f;
d_out[index] = d_in[index];
__syncthreads();

for (int i = 1; i < sizeof(d_in); i*=2)
{
if (index - i >= 0)
{
out = d_out[index] + d_out[index - i];
}
__syncthreads();

if (index - i >=0)
{
d_out[index] = out;
out = 0.0f;
}
}

}