#include "hip/hip_runtime.h"
#include "includes.h"



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);





__global__ void shmem_scan(float* d_out, float* d_in) {
extern __shared__ float sdata[];
int idx = threadIdx.x;
float out = 0.00f;
sdata[idx] = d_in[idx];
__syncthreads();
for (int interpre = 1; interpre < sizeof(d_in); interpre *= 2) {
if (idx - interpre >= 0) {
out = sdata[idx] + sdata[idx - interpre];
}
__syncthreads();
if (idx - interpre >= 0) {
sdata[idx] = out;
out = 0.00f;
}
}
d_out[idx] = sdata[idx];
}