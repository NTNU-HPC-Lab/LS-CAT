#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelF(const float *d_x, float *d_y)
{
const float &x0 = d_x[0];
const float &x1 = d_x[1];

// f = (1-x0)^2 + 100 (x1-x0^2)^2

const float a = (1.0 - x0);
const float b = (x1 - x0 * x0) ;

*d_y = (a*a) + 100.0f * (b*b);
}