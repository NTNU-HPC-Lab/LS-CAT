#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_pillar_feature_kernel( float* dev_pillar_x_in_coors, float* dev_pillar_y_in_coors, float* dev_pillar_z_in_coors, float* dev_pillar_i_in_coors, float* dev_pillar_x, float* dev_pillar_y, float* dev_pillar_z, float* dev_pillar_i, int* dev_x_coors, int* dev_y_coors, float* dev_num_points_per_pillar, const int max_points, const int GRID_X_SIZE)
{
int ith_pillar = blockIdx.x;
int num_points_at_this_pillar = dev_num_points_per_pillar[ith_pillar];
int ith_point = threadIdx.x;
if(ith_point >= num_points_at_this_pillar)
{
return;
}
int x_ind = dev_x_coors[ith_pillar];
int y_ind = dev_y_coors[ith_pillar];
int pillar_ind = ith_pillar*max_points + ith_point;
int coors_ind = y_ind*GRID_X_SIZE*max_points + x_ind*max_points + ith_point;
dev_pillar_x[pillar_ind] = dev_pillar_x_in_coors[coors_ind];
dev_pillar_y[pillar_ind] = dev_pillar_y_in_coors[coors_ind];
dev_pillar_z[pillar_ind] = dev_pillar_z_in_coors[coors_ind];
dev_pillar_i[pillar_ind] = dev_pillar_i_in_coors[coors_ind];
}