#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
float t = truth[i];
float p = pred[i];
error[i] = -t*log(p+.0000001) - (1-t)*log(1-p+.0000001);
delta[i] = t-p;
}
}