#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumScore(double *score, int full_size, int half_size)
{
int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = index; i < half_size; i += stride) {
score[i] += (i + half_size < full_size) ? score[i + half_size] : 0;
}
}