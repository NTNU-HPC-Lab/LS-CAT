#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "sort_boxes_by_indexes_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *filtered_box = NULL;
hipMalloc(&filtered_box, XSIZE*YSIZE);
int *filtered_dir = NULL;
hipMalloc(&filtered_dir, XSIZE*YSIZE);
float *box_for_nms = NULL;
hipMalloc(&box_for_nms, XSIZE*YSIZE);
int *indexes = NULL;
hipMalloc(&indexes, XSIZE*YSIZE);
int filter_count = 2;
float *sorted_filtered_boxes = NULL;
hipMalloc(&sorted_filtered_boxes, XSIZE*YSIZE);
int *sorted_filtered_dir = NULL;
hipMalloc(&sorted_filtered_dir, XSIZE*YSIZE);
float *sorted_box_for_nms = NULL;
hipMalloc(&sorted_box_for_nms, XSIZE*YSIZE);
const int NUM_BOX_CORNERS = 1;
const int NUM_OUTPUT_BOX_FEATURE = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sort_boxes_by_indexes_kernel<<<gridBlock,threadBlock>>>(filtered_box,filtered_dir,box_for_nms,indexes,filter_count,sorted_filtered_boxes,sorted_filtered_dir,sorted_box_for_nms,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
sort_boxes_by_indexes_kernel<<<gridBlock,threadBlock>>>(filtered_box,filtered_dir,box_for_nms,indexes,filter_count,sorted_filtered_boxes,sorted_filtered_dir,sorted_box_for_nms,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
sort_boxes_by_indexes_kernel<<<gridBlock,threadBlock>>>(filtered_box,filtered_dir,box_for_nms,indexes,filter_count,sorted_filtered_boxes,sorted_filtered_dir,sorted_box_for_nms,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}