#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_pillar_histo_kernel( const float* dev_points, float* dev_pillar_x_in_coors, float* dev_pillar_y_in_coors, float* dev_pillar_z_in_coors, float* dev_pillar_i_in_coors, int* pillar_count_histo, const int num_points, const int max_points_per_pillar, const int GRID_X_SIZE, const int GRID_Y_SIZE, const int GRID_Z_SIZE, const float MIN_X_RANGE, const float MIN_Y_RANGE, const float MIN_Z_RANGE, const float PILLAR_X_SIZE, const float PILLAR_Y_SIZE, const float PILLAR_Z_SIZE, const int NUM_BOX_CORNERS )
{
int th_i = threadIdx.x + blockIdx.x * blockDim.x;
if(th_i >= num_points)
{
return;
}
int y_coor = floor((dev_points[th_i*NUM_BOX_CORNERS + 1] - MIN_Y_RANGE)/PILLAR_Y_SIZE);
int x_coor = floor((dev_points[th_i*NUM_BOX_CORNERS + 0] - MIN_X_RANGE)/PILLAR_X_SIZE);
int z_coor = floor((dev_points[th_i*NUM_BOX_CORNERS + 2] - MIN_Z_RANGE)/PILLAR_Z_SIZE);

if(x_coor >= 0 && x_coor < GRID_X_SIZE &&
y_coor >= 0 && y_coor < GRID_Y_SIZE &&
z_coor >= 0 && z_coor < GRID_Z_SIZE)
{
int count = atomicAdd(&pillar_count_histo[y_coor*GRID_X_SIZE + x_coor], 1);
if(count < max_points_per_pillar)
{
int ind = y_coor*GRID_X_SIZE*max_points_per_pillar + x_coor*max_points_per_pillar + count;
dev_pillar_x_in_coors[ind] = dev_points[th_i*NUM_BOX_CORNERS + 0];
dev_pillar_y_in_coors[ind] = dev_points[th_i*NUM_BOX_CORNERS + 1];
dev_pillar_z_in_coors[ind] = dev_points[th_i*NUM_BOX_CORNERS + 2];
dev_pillar_i_in_coors[ind] = dev_points[th_i*NUM_BOX_CORNERS + 3];
}
}
}