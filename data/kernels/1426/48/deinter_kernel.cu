#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < (NX+NY)*B){
int b = i / (NX+NY);
int j = i % (NX+NY);
if (j < NX){
if(X) X[b*NX + j] += OUT[i];
} else {
if(Y) Y[b*NY + j - NX] += OUT[i];
}
}
}