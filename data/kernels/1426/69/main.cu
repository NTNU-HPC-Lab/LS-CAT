#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "filter_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *box_preds = NULL;
hipMalloc(&box_preds, XSIZE*YSIZE);
const float *cls_preds = NULL;
hipMalloc(&cls_preds, XSIZE*YSIZE);
const float *dir_preds = NULL;
hipMalloc(&dir_preds, XSIZE*YSIZE);
const int *anchor_mask = NULL;
hipMalloc(&anchor_mask, XSIZE*YSIZE);
const float *dev_anchors_px = NULL;
hipMalloc(&dev_anchors_px, XSIZE*YSIZE);
const float *dev_anchors_py = NULL;
hipMalloc(&dev_anchors_py, XSIZE*YSIZE);
const float *dev_anchors_pz = NULL;
hipMalloc(&dev_anchors_pz, XSIZE*YSIZE);
const float *dev_anchors_dx = NULL;
hipMalloc(&dev_anchors_dx, XSIZE*YSIZE);
const float *dev_anchors_dy = NULL;
hipMalloc(&dev_anchors_dy, XSIZE*YSIZE);
const float *dev_anchors_dz = NULL;
hipMalloc(&dev_anchors_dz, XSIZE*YSIZE);
const float *dev_anchors_ro = NULL;
hipMalloc(&dev_anchors_ro, XSIZE*YSIZE);
float *filtered_box = NULL;
hipMalloc(&filtered_box, XSIZE*YSIZE);
float *filtered_score = NULL;
hipMalloc(&filtered_score, XSIZE*YSIZE);
int *filtered_dir = NULL;
hipMalloc(&filtered_dir, XSIZE*YSIZE);
float *box_for_nms = NULL;
hipMalloc(&box_for_nms, XSIZE*YSIZE);
int *filter_count = NULL;
hipMalloc(&filter_count, XSIZE*YSIZE);
const float FLOAT_MIN = 1;
const float FLOAT_MAX = 1;
const float score_threshold = 1;
const int NUM_BOX_CORNERS = 1;
const int NUM_OUTPUT_BOX_FEATURE = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
filter_kernel<<<gridBlock,threadBlock>>>(box_preds,cls_preds,dir_preds,anchor_mask,dev_anchors_px,dev_anchors_py,dev_anchors_pz,dev_anchors_dx,dev_anchors_dy,dev_anchors_dz,dev_anchors_ro,filtered_box,filtered_score,filtered_dir,box_for_nms,filter_count,FLOAT_MIN,FLOAT_MAX,score_threshold,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
filter_kernel<<<gridBlock,threadBlock>>>(box_preds,cls_preds,dir_preds,anchor_mask,dev_anchors_px,dev_anchors_py,dev_anchors_pz,dev_anchors_dx,dev_anchors_dy,dev_anchors_dz,dev_anchors_ro,filtered_box,filtered_score,filtered_dir,box_for_nms,filter_count,FLOAT_MIN,FLOAT_MAX,score_threshold,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
filter_kernel<<<gridBlock,threadBlock>>>(box_preds,cls_preds,dir_preds,anchor_mask,dev_anchors_px,dev_anchors_py,dev_anchors_pz,dev_anchors_dx,dev_anchors_dy,dev_anchors_dz,dev_anchors_ro,filtered_box,filtered_score,filtered_dir,box_for_nms,filter_count,FLOAT_MIN,FLOAT_MAX,score_threshold,NUM_BOX_CORNERS,NUM_OUTPUT_BOX_FEATURE);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}