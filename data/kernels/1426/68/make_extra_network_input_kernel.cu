#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_extra_network_input_kernel(float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, float* dev_num_points_per_pillar, float* dev_x_coors_for_sub_shaped, float* dev_y_coors_for_sub_shaped, float* dev_pillar_feature_mask, const int MAX_NUM_POINTS_PER_PILLAR)
{
int ith_pillar = blockIdx.x;
int ith_point = threadIdx.x;
float x = dev_x_coors_for_sub[ith_pillar];
float y = dev_y_coors_for_sub[ith_pillar];
int num_points_for_a_pillar = dev_num_points_per_pillar[ith_pillar];
int ind = ith_pillar*MAX_NUM_POINTS_PER_PILLAR + ith_point;
dev_x_coors_for_sub_shaped[ind] = x;
dev_y_coors_for_sub_shaped[ind] = y;

if(ith_point < num_points_for_a_pillar)
{
dev_pillar_feature_mask[ind] = 1.0;
}
else
{
dev_pillar_feature_mask[ind] = 0.0;
}
}