#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knapsackGPU(int* dp, int row, int* d_value, int* d_weight,int capacity)
{
int in = threadIdx.x + (blockDim.x * blockIdx.x);
if (row != 0)
{
int ind = in + (row * (capacity+1));
if (in <= (capacity+1) && in > 0)
{
if (in >= d_weight[row - 1])
{
dp[ind] = dp[ind - (capacity+1)]> (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]) ? dp[ind - (capacity + 1)] : (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]);
}
else
dp[ind] = dp[ind - (capacity+1)];
}
if (in == 0)
{
dp[ind] = 0;
}
}
else
{
dp[in] = 0;
}
}