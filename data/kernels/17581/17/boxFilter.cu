#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void boxFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// only threads inside image will write results
if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
{
for(int c=0 ; c<channel ; c++)
{
// Sum of pixel values
float sum = 0;
// Number of filter pixels
float kS = 0;
// Loop inside the filter to average pixel values
for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
float fl = srcImage[((y+ky)*width + (x+kx))*channel+c];
sum += fl;
kS += 1;
}
}
dstImage[(y*width+x)*channel+c] =  sum / kS;
}
}
}