#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void sort(unsigned char* filterVector)
{
for (int i = 0; i < FILTER_WIDTH*FILTER_HEIGHT; i++) {
for (int j = i + 1; j < FILTER_WIDTH*FILTER_HEIGHT; j++) {
if (filterVector[i] > filterVector[j]) {
//Swap the variables
unsigned char tmp = filterVector[i];
filterVector[i] = filterVector[j];
filterVector[j] = tmp;
}
}
}
}
__global__ void medianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// only threads inside image will write results
if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
{
for(int c=0 ; c<channel ; c++)
{
unsigned char filterVector[FILTER_WIDTH*FILTER_HEIGHT];
// Loop inside the filter to average pixel values
for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
filterVector[ky*FILTER_WIDTH+kx] = srcImage[((y+ky)*width + (x+kx))*channel+c];
}
}
// Sorting values of filter
sort(filterVector);
dstImage[(y*width+x)*channel+c] =  filterVector[(FILTER_WIDTH*FILTER_HEIGHT)/2];
}
}
}