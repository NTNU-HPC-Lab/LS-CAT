#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sharpeningFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

float kernel[FILTER_WIDTH][FILTER_HEIGHT] = { {-1, -1, -1}, {-1, 9, -1}, {-1, -1, -1} };
// only threads inside image will write results
if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
{
for (int c = 0; c < channel; c++)
{
// Sum of pixel values
float sum = 0;
// Loop inside the filter to average pixel values
for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
}
}
dstImage[(y * width + x) * channel + c] = sum;
}
}
}