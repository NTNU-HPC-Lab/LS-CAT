#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void laplacianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

float ker[3][3] = {{0, -1, 0}, {-1, 4, -1}, {0, -1, 0}};
//float kernel[3][3] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};
// only threads inside image will write results
if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
{
// Sum of pixel values
float sum = 0;
// Loop inside the filter to average pixel values
for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
float fl = srcImage[((y+ky)*width + (x+kx))];
sum += fl*ker[ky+FILTER_HEIGHT/2][kx+FILTER_WIDTH/2];
}
}
dstImage[(y*width+x)] =  sum;
}
}