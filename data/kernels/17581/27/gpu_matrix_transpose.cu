#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

if (idx < cols && idy < rows)
{
unsigned int pos = idy * cols + idx;
unsigned int trans_pos = idx * rows + idy;
mat_out[trans_pos] = mat_in[pos];
}
}