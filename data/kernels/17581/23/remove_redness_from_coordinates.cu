#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void remove_redness_from_coordinates( const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int    num_coordinates, int    num_pixels_y, int    num_pixels_x, int    template_half_height, int    template_half_width )
{
int ny = num_pixels_y;
int nx = num_pixels_x;
int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

int imgSize = num_pixels_x * num_pixels_y;

if (global_index_1d < num_coordinates)
{
unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
{
for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
{
int2 image_offset_index_2d = make_int2(x, y);
int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

unsigned char g_value = d_g[image_offset_index_1d_clamped];
unsigned char b_value = d_b[image_offset_index_1d_clamped];

unsigned int gb_average = (g_value + b_value) / 2;
//printf("heya\t");
d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
}
}
}
}