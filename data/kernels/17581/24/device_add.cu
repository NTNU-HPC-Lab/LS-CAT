#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_add(int* a,int* b,int* c)
{
int index = threadIdx.x + blockIdx.x*blockDim.x;
c[index] = a[index] + b[index];
}