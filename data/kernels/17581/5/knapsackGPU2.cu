#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knapsackGPU2(int* dp, int* d_value, int* d_weight, int capacity,int n)
{
int in = threadIdx.x + (blockDim.x * blockIdx.x);
for (int row = 0;row <= n;row++)
{
if (row != 0)
{
int ind = in + (row * (capacity + 1));
if (in <= (capacity + 1) && in > 0)
{
if (in >= d_weight[row - 1])
{
dp[ind] = dp[ind - (capacity + 1)] > (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]) ? dp[ind - (capacity + 1)] : (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]);
}
else
dp[ind] = dp[ind - (capacity + 1)];
}
if (in == 0)
{
dp[ind] = 0;
}
}
else
{
dp[in] = 0;
}
}

}