#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
int  ny             = num_pixels_y;
int  nx             = num_pixels_x;
int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

if ( image_index_2d.x < nx && image_index_2d.y < ny )
{
float x         = d_x[ image_index_1d ];
float y         = d_y[ image_index_1d ];
float log_Y     = d_log_Y[ image_index_1d ];
int   bin_index = min( num_bins - 1, int( (num_bins * ( log_Y - min_log_Y ) ) / log_Y_range ) );
float Y_new     = d_cdf_norm[ bin_index ];

float X_new = x * ( Y_new / y );
float Z_new = ( 1 - x - y ) * ( Y_new / y );

float r_new = ( X_new *  3.2406f ) + ( Y_new * -1.5372f ) + ( Z_new * -0.4986f );
float g_new = ( X_new * -0.9689f ) + ( Y_new *  1.8758f ) + ( Z_new *  0.0415f );
float b_new = ( X_new *  0.0557f ) + ( Y_new * -0.2040f ) + ( Z_new *  1.0570f );

d_r_new[ image_index_1d ] = r_new;
d_g_new[ image_index_1d ] = g_new;
d_b_new[ image_index_1d ] = b_new;
}
}