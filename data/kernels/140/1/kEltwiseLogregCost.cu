#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kEltwiseLogregCost(float* predmap, float* indmap, float*indlogpred, float* correctprobs, int numCases, int numTasks, int per_thread_case) {
const int task_id = blockIdx.x;
const int start_tx = threadIdx.x * per_thread_case;
const int end_tx = min(start_tx + per_thread_case, numCases);
const float EPSILON=1e-20; // Minimum value allowed, avoid log( 0 )
if (task_id >= numTasks) {
return;
}
for (int c_id = start_tx; c_id < end_tx; ++c_id) {
int pos = task_id * numCases + c_id;
float t = __fdividef(1.0f, 1.0f + __expf(-predmap[ pos ]));
if (indmap[pos] == 1) {
t = fmaxf(t, EPSILON);
indlogpred[pos] = __logf(t);
correctprobs[pos] = t;
} else {
t = 1-t;
t = fmaxf(t, EPSILON);
indlogpred[pos] = __logf(t);
correctprobs[pos] = t;
}
}
}