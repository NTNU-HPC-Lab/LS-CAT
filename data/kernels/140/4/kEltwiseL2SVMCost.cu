#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kEltwiseL2SVMCost(float* ydata, float* ldata, float* pre_grad, float* all_cost, float a, float b, int numCases, int numTasks, int per_thread_case) {
const int task_id = blockIdx.x;
const int start_tx = threadIdx.x * per_thread_case;
const int end_tx = min(start_tx + per_thread_case, numCases);
if (task_id >= numTasks) {
return;
}
for (int c_id = start_tx; c_id < end_tx; ++c_id) {
int pos = task_id * numCases + c_id;
float tmp = fmaxf(a - ydata[pos] * (ldata[pos] - b), 0);
pre_grad[pos] = tmp;
all_cost[pos] = tmp*tmp;
}
}