#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned int*) end_ptr) {
return;
}


for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i++){
ptr[i] = pattern;
}

return;
}