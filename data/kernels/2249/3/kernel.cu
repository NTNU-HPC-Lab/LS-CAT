#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel()
{
/*
this just gets some kernel specific parameters
this is just so you can see how non-deterministic thread timing is
*/
int tidx = threadIdx.x + blockIdx.x * blockDim.x;
int tidy = threadIdx.y + blockIdx.y * blockDim.y;

/* print some stuff out */
int size = sizeof(int);
printf("Hello, World! size=%d   tidx=%d, tidy=%d\n", size, tidx, tidy);
return;
}