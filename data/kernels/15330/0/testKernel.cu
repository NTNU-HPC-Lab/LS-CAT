#include "hip/hip_runtime.h"
#include "includes.h"

// This works fine with a mutex, but crashes with a sigbus error when not using a mutex
// #define USE_MUTEX

#ifdef USE_MUTEX
std::mutex m;
#endif


__global__ void testKernel() {
printf("Thread Kernel running\n");
}