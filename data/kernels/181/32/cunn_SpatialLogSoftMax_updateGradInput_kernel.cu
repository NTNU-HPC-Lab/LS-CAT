#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int feature_size, int spatial_size, int data_size)
{
int idx = (threadIdx.x + blockDim.x*blockIdx.x);
idx = (idx/spatial_size)*feature_size + idx % spatial_size;

if (idx < data_size) {
int next_idx = idx + feature_size;
float gradSum = 0.0;
// Compute the sum of gradients
for(int i = idx; i < next_idx; i += spatial_size){
gradSum += gradOutput[i];
}
// Compute the new gradient
for(int i = idx; i < next_idx; i += spatial_size){
gradInput[i] = gradOutput[i] - __expf(output[i])*gradSum;
}
}
}