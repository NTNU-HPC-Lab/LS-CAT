#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;




__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size)
{
int di = blockIdx.x * block_size + threadIdx.x;
dst[di] = src1[di] * src2[di];
}