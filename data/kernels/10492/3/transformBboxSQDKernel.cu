#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;




__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, int block_size)
{
int di = (blockIdx.x * block_size + threadIdx.x) * 4;
float d[4] = {delta[di], delta[di+1], delta[di+2], delta[di+3]};
float a[4] = {anchor[di], anchor[di+1], anchor[di+2], anchor[di+3]};
float cx = a[0] + d[0] * a[2];
float cy = a[1] + d[1] * a[3];
float w = a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E);
float h = a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E);
res[di] = cx - w * 0.5;
res[di+1] = cy - h * 0.5;
res[di+2] = cx + w * 0.5;
res[di+3] = cy + h * 0.5;
}