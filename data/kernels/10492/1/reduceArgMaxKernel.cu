#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;




__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int block_size)
{
int di = blockIdx.x * block_size + threadIdx.x;
int si = di * dim_size;
float now = src[si], max = now;
int maxi = 0;
for (int i = 1; i < dim_size; i++) {
now = src[si+i];
if (now > max) {
max = now;
maxi = i;
}
}
dst[di] = max;
arg[di] = maxi;
}