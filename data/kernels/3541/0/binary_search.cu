#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void binary_search(int* a, int* b, bool* c, int sizeofa) //kernal function
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
printf(" %d\n", index);
int key = b[index];
int min = 0, max = sizeofa;
int mid = sizeofa / 2;
while (min != mid)
{
if (key == a[mid])
{
break;
}
else if (key < a[mid])
{
min = min;
max = mid;
}
else {
min = mid;
max = max;
}
mid = (min + max) / 2;
}

if (key == a[mid])
c[index] = true;
else
c[index] = false;

printf(" %d %d  %d %d\n", index, key, a[mid],c[index]);
}