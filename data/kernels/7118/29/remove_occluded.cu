#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void remove_occluded(float *y, int size, int size3)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int x = id % size3;
for (int i = 1; x + i < size3; i++) {
if (i - y[id + i] < -y[id]) {
y[id] = 0;
break;
}
}
}
}