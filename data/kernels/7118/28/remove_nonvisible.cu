#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void remove_nonvisible(float *y, int size, int size3)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int x = id % size3;
if (y[id] >= x) {
y[id] = 0;
}
}
}