#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int out_x = id % out_size3;
int out_y = id / out_size3;

int in_x = out_x - (out_size3 - in_size3) / 2;
int in_y = out_y - (out_size2 - in_size2) / 2;

int x = min(in_size3 - 1, max(0, in_x));
int y = min(in_size2 - 1, max(0, in_y));

out[id] = in[y * in_size3 + x];
}
}