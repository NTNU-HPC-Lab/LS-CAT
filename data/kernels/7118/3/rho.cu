#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
x[id] = 1 - exp(-x[id] / lambda);
}
}