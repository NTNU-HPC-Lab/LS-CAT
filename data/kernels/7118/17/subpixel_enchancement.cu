#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int d = d0[id];
out[id] = d;
if (1 <= d && d < disp_max - 1) {
float cn = c2[(d - 1) * dim23 + id];
float cz = c2[d * dim23 + id];
float cp = c2[(d + 1) * dim23 + id];
float denom = 2 * (cp + cn - 2 * cz);
if (denom > 1e-5) {
out[id] = d - min(1.0, max(-1.0, (cp - cn) / denom));
}
}
}
}