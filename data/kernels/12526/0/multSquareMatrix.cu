#include "hip/hip_runtime.h"
#include "includes.h"




using namespace std;





#define N 32

__global__ void multSquareMatrix(int *A, int *B, int *result, int n)
{
int k, sum = 0;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

for (k = 0; k < n; k++) {
sum += A[row * n + k] * B[k * n + col];
result[row * n + col] = sum;
}
}