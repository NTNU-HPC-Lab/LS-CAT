#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_shared( int *A, int *B, int *result, int n)
{	int k;
int kk;
const int bx = BLOCK_X, by = BLOCK_Y;
const int col = blockIdx.x*bx + threadIdx.x;
const int row = blockIdx.y*by + threadIdx.y;

__shared__ int a[BLOCK_X][BLOCK_Y] , b[BLOCK_X][BLOCK_Y];
if ((col < n) && (row < n))
{
int c = 0;
for (k=0; k < n; k++)
{
a[threadIdx.x][threadIdx.y] = A[ col * n + k*by + threadIdx.y];
b[threadIdx.y][threadIdx.x] = B[ row + n * (k*bx+threadIdx.x)];
__syncthreads(); // Synchronizes all threads in a block
for (kk=0; kk< bx; kk++)
c += a[kk][threadIdx.x]*b[kk][threadIdx.y];
__syncthreads(); // Avoids memory hazards
}
result[col*n+row] = c;
}

}