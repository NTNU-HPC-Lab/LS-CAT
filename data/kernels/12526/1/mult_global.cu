#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_global (int *A, int *B, int *result, int n)
{
int k, sum = 0;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

if(col < n && row  < n)
{
for (k = 0; k < n; k++)
{
sum += A[row * n + k] * B[k * n + col];
result[row * n + col] = sum;
}
}
}