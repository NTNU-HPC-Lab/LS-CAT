#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void replaceNaNKernel( int numberEntriesPerInstance, int numberIterations, float* source, float* destination) {

int instanceIndex = blockIdx.x;

int instanceStart = instanceIndex * numberEntriesPerInstance;
int startWithinThread = instanceStart + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

for(int index = startWithinThread; index < min(startWithinThread + numberIterations, instanceStart + numberEntriesPerInstance); index++) {
float currentValue = source[index];
destination[index] = isnan(currentValue) ? 0.0f : currentValue;
}
}