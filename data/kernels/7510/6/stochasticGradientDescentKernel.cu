#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stochasticGradientDescentKernel ( int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate) {

int updateIndex = blockIdx.x;
int parameterIndex = parameterIndices[updateIndex];
int count = counts[updateIndex];

if(parameterIndex != -1 && count > 0) {

float scalingFactor = 1.0f / (float)count;

int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

int firstParameterEntryIndex = parameterIndex * dimension;
int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

int parameterEntryIndex = startParameterEntryIndex;
int gradientEntryIndex = startGradientEntryIndex;

while(parameterEntryIndex < exclusiveEndParameterEntryIndex) {
float scaledDerivative = scalingFactor * gradient[gradientEntryIndex];

parameters[parameterEntryIndex] -= learningRate * scaledDerivative;

parameterEntryIndex++;
gradientEntryIndex++;
}
}
}