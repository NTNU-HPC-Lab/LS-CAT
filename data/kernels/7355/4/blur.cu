#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blur( float * input, float * output, int  height, int width)
{

int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if(x<height && y<width)
{
for(int k=0;k<3;k++)
{
float sum=0;
int count=0;
for(int i=x-BLUR_SIZE; i<= x+BLUR_SIZE; i++)
{
for(int j= y-BLUR_SIZE; j<=y+BLUR_SIZE;j++)
{
if(i>=0 && i<height && j>=0 && j<width)
{
count++;
sum+=input[3*(i*width+j)+k];
}
}
}
output[3*(x*width+y)+k]=sum/count;
}
}
else
return ;
}