#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2gray (float * input, float *output, int height, int width)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if(x<height && y<width)
{
unsigned int idx = x* width + y;
float r          = input[3 * idx];
float g          = input[3 * idx + 1]; // green value for pixel
float b          = input[3 * idx + 2];
output[idx] = (0.21f * r + 0.71f * g + 0.07f * b);
}
}