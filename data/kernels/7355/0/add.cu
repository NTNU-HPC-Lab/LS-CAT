#include "hip/hip_runtime.h"
#include "includes.h"

# define M 10000
# define N 10000


__global__ void add( int * a, int * b, int * c)
{
unsigned int i= blockDim.x *blockIdx.x + threadIdx.x;
unsigned int j= blockDim.y *blockIdx.y + threadIdx.y;
if(i<M && j<N)
c[i*M+j]=a[i*M+j]+b[i*M+j];
}