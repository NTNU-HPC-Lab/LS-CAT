#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double atomicAdd_dB(double* address, double val)
{
unsigned long long int* address_as_ull =
(unsigned long long int*)address;
unsigned long long int old = *address_as_ull, assumed;

do {
assumed = old;
old = atomicCAS(address_as_ull, assumed,
__double_as_longlong(val +
__longlong_as_double(assumed)));

// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
} while (assumed != old);

return __longlong_as_double(old);
}
__device__ double atomicAdd_d(double* address, double val)
{
unsigned long long int* address_as_ull =
(unsigned long long int*)address;
unsigned long long int old = *address_as_ull, assumed;

do {
assumed = old;
old = atomicCAS(address_as_ull, assumed,
__double_as_longlong(val +
__longlong_as_double(assumed)));

// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
} while (assumed != old);

return __longlong_as_double(old);
}
__global__ void kennel_matrixQ(double *d_P, double *d_px, double *d_py, double *d_Q, int Ng)
{
int rownum,colnum;
int tid = threadIdx.x + blockDim.x * blockIdx.x;
double temp;

rownum = tid/Ng;
colnum = tid%Ng;
d_Q[(tid+Ng)]=0;	//Ng is added to point in next row

for(int k=0; k<Ng; ++k)
{
if (d_px[rownum] && d_py[k])  // make sure to protect division by zero
{
temp=d_P[k+rownum*Ng]*d_P[k+colnum*Ng]/d_px[rownum]/d_py[k];
atomicAdd_dB(&d_Q[(tid+Ng)],temp);
}
}
}