#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process_coarseness_ek_pix(double * output_ak, double *output_ekh, double *output_ekv,int colsize, int rowsize,long lenOf_ek)
{
int y  = threadIdx.x + blockIdx.x * blockDim.x;
int x = threadIdx.y + blockIdx.y * blockDim.y;
double input1,input2;
int posx1 = x+lenOf_ek;
int posx2 = x-lenOf_ek;
int posy1 = y+lenOf_ek;
int posy2 = y-lenOf_ek;
if(y < (colsize) && x < (rowsize))
{
if(posx1 < (int)rowsize && posx2 >= 0)
{
input1 = output_ak[y * rowsize + posx1];
input2 = output_ak[y * rowsize + posx2];
output_ekh[y*rowsize+x] = fabs(input1 - input2);
}
else output_ekh[y*rowsize+x] = 0;

if(posy1 < (int)colsize && posy2 >= 0)
{
input1 = output_ak[posy1 * rowsize + x];
input2 = output_ak[posy2 * rowsize + x];
output_ekv[y*rowsize+x] = fabs(input1 - input2);
}
else output_ekv[y*rowsize+x] = 0;
}
}