#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double efficientLocalMean_dev (const long x,const long y,const long k, double * input_img, int rowsize, int colsize) {
long k2 = k/2;

long dimx = rowsize;
long dimy = colsize;

//wanting average over area: (y-k2,x-k2) ... (y+k2-1, x+k2-1)
long starty = y-k2;
long startx = x-k2;
long stopy = y+k2-1;
long stopx = x+k2-1;

if (starty < 0) starty = 0;
if (startx < 0) startx = 0;
if (stopx > dimx-1) stopx = dimx-1;
if (stopy > dimy-1) stopy = dimy-1;

double unten, links, oben, obenlinks;

if (startx-1 < 0) links = 0;
else links = *(input_img+(stopy * dimx + startx-1));

if (starty-1 < 0) oben = 0;
else oben = *(input_img+((stopy-1) * dimx + startx));

if ((starty-1 < 0) || (startx-1 <0)) obenlinks = 0;
else obenlinks = *(input_img+((stopy-1) * dimx + startx-1));

unten = *(input_img+(stopy * dimx + startx));

long counter = (stopy-starty+1)*(stopx-startx+1);
return (unten-links-oben+obenlinks)/counter;
}
__global__ void process_coarseness_ak_pix(double * output_ak,double * input_img,int colsize, int rowsize,long lenOf_ak)
{
int index;
int y  = threadIdx.x + blockIdx.x * blockDim.x;
int x = threadIdx.y + blockIdx.y * blockDim.y;
if(y < (colsize) && x < (rowsize))
{
index = y * rowsize + x ;
output_ak[index] = efficientLocalMean_dev(x,y,lenOf_ak,input_img,rowsize,colsize);
}
}