#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void compute_conv(int row, int col, double2 *d_c, double *d_a, double2 *d_b, int *o_row_vect, int *o_col_vect, int ma, int na, int mb, int nb, int mc, int nc) {

int count_row = o_row_vect[row];
int count_col = o_col_vect[col];
int row_idx;
int col_idx;
int k_row_idx;
int k_col_idx;
int k_col_start_idx;

int i_row_idx;
int i_col_idx;
int i_col_start_idx;

k_row_idx = row - (ma - 1);
k_row_idx = k_row_idx < 0 ? 0:k_row_idx;

k_col_start_idx = col - (na - 1);
k_col_start_idx = k_col_start_idx < 1? 0: k_col_start_idx;
k_col_idx = k_col_start_idx;

i_row_idx = row > (ma - 1) ? (ma - 1) : row;
i_col_idx = col > (na - 1) ? (na - 1) : col;
i_col_start_idx = i_col_idx;

for ( row_idx = 0; row_idx < count_row; row_idx++) {
for (col_idx = 0; col_idx < count_col; col_idx++) {

d_c[col + nc * row].x += d_a[i_col_idx + na * i_row_idx] * d_b[k_col_idx + nb * k_row_idx].x;
d_c[col + nc * row].y += d_a[i_col_idx + na * i_row_idx] * d_b[k_col_idx + nb * k_row_idx].y;

k_col_idx++;
i_col_idx--;
}
k_row_idx++;
i_row_idx--;
k_col_idx = k_col_start_idx;
i_col_idx = i_col_start_idx;

}
}
__global__ void kernel_conv(double2 *d_c, double *d_a, double2 *d_b, int *d_row_vect, int *d_col_vect, int ma, int na, int mb, int nb, int mc, int nc) {

int i, idx;
int rownum, colnum, num_threads;

idx = threadIdx.x + blockIdx.x * blockDim.x;
num_threads = gridDim.x * blockDim.x;

for(i=idx; i< (mc *nc); i=i+num_threads){

rownum = i / nc;
colnum = i % nc;

// Device Function call to multiply the Image pixel with the Kernel Image pixel and perform addition
compute_conv(rownum, colnum, d_c, d_a, d_b, d_row_vect, d_col_vect, ma, na, mb, nb, mc, nc);
}
}