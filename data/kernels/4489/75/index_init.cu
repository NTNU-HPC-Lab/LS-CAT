#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void index_init(int* out_data, int h, int w) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
for (int i = idx; i < h * w; i += blockDim.x * gridDim.x) {
int w_id = i % w;
out_data[i] = w_id;
}
}