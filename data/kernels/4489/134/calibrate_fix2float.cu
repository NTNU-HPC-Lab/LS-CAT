#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calibrate_fix2float(float * dst, const float* sA, const float* sB, float alpha, float beta, int height, int width, int threads) {
int ri = blockIdx.x;
int tid = threadIdx.x;
int loop = (width / threads) + ((width % threads == 0) ? 0 : 1);

float rscale = (sA[ri] == 0.0f) ? 1.0f : sA[ri];
float * data = dst + width * ri;
int idx = 0;
for (int i = 0; i < loop; ++i) {
if(idx + tid < width){
float temp = data[idx + tid];
float cscale = (sB[idx + tid] == 0.0f) ? 255.0f : sB[idx + tid];
data[idx + tid] = beta  * temp + alpha * temp * rscale * cscale;
}
idx += threads;
}
}