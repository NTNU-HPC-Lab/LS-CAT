#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "__pairmult2.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int nrows = 1;
int bncols = 1;
int brows1 = 1;
int brows2 = 1;
float *A = NULL;
hipMalloc(&A, XSIZE*YSIZE);
int lda = 1;
float *A2 = NULL;
hipMalloc(&A2, XSIZE*YSIZE);
int lda2 = 1;
float *Bdata = NULL;
hipMalloc(&Bdata, XSIZE*YSIZE);
int *Bir = NULL;
hipMalloc(&Bir, XSIZE*YSIZE);
int *Bjc = NULL;
hipMalloc(&Bjc, XSIZE*YSIZE);
int broff = 1;
int bcoff = 1;
float *C = NULL;
hipMalloc(&C, XSIZE*YSIZE);
int ldc = 1;
int transpose = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
__pairmult2<<<gridBlock,threadBlock>>>(nrows,bncols,brows1,brows2,A,lda,A2,lda2,Bdata,Bir,Bjc,broff,bcoff,C,ldc,transpose);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
__pairmult2<<<gridBlock,threadBlock>>>(nrows,bncols,brows1,brows2,A,lda,A2,lda2,Bdata,Bir,Bjc,broff,bcoff,C,ldc,transpose);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
__pairmult2<<<gridBlock,threadBlock>>>(nrows,bncols,brows1,brows2,A,lda,A2,lda2,Bdata,Bir,Bjc,broff,bcoff,C,ldc,transpose);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}