#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __findBoundaries(long long *keys, int *jc, int n, int njc, int shift) {
__shared__ int dbuff[1024];
int i, j, iv, lasti;

int imin = ((int)(32 * ((((long long)n) * blockIdx.x) / (gridDim.x * 32))));
int imax = min(n, ((int)(32 * ((((long long)n) * (blockIdx.x + 1)) / (gridDim.x * 32) + 1))));

int tid = threadIdx.x + blockDim.x * threadIdx.y;
if (tid == 0 && blockIdx.x == 0) {
jc[0] = 0;
}
__syncthreads();
lasti = 0x7fffffff;
for (i = imin; i <= imax; i += blockDim.x * blockDim.y) {
iv = njc;
if (i + tid < imax) {
iv = (int)(keys[i + tid] >> shift);
dbuff[tid] = iv;
}
__syncthreads();
if (i + tid < imax || i + tid == n) {
if (tid > 0) lasti = dbuff[tid - 1];
if (iv > lasti) {
for (j = lasti+1; j <= iv; j++) {
jc[j] = i + tid;
}
}
if (tid == 0) {
lasti = dbuff[blockDim.x * blockDim.y - 1];
}
}
__syncthreads();
}
}