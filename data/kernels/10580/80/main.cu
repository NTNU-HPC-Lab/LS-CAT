#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "__ADAGrad.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int nrows = 1;
int ncols = 1;
float *mm = NULL;
hipMalloc(&mm, XSIZE*YSIZE);
float *um = NULL;
hipMalloc(&um, XSIZE*YSIZE);
float *ssq = NULL;
hipMalloc(&ssq, XSIZE*YSIZE);
float *mask = NULL;
hipMalloc(&mask, XSIZE*YSIZE);
int maskr = 1;
float nw = 1;
float *ve = NULL;
hipMalloc(&ve, XSIZE*YSIZE);
int nve = 1;
float *ts = NULL;
hipMalloc(&ts, XSIZE*YSIZE);
int nts = 1;
float *lr = NULL;
hipMalloc(&lr, XSIZE*YSIZE);
int nlr = 1;
float langevin = 1;
float eps = 1;
int doupdate = 1;
hiprandState *rstates = NULL;
hipMalloc(&rstates, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
__ADAGrad<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
__ADAGrad<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
__ADAGrad<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}