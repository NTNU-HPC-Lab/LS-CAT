#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __ADAGrad(int nrows, int ncols, float *mm, float *um, float *ssq, float *mask, int maskr, float nw, float *ve, int nve, float *ts, int nts, float *lr, int nlr, float langevin, float eps, int doupdate, hiprandState *rstates) {
int ithread = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int nthreads = blockDim.x * gridDim.x * gridDim.y;
int i, irow, icol;
float mmval, umval, sqrtss, sqrtnewss, veval, tsval, lrval, denom, grad;
float sqrtnw = sqrtf(nw);
float sqrt1mnw = sqrtf(1-nw);
float sqrteps = sqrt(eps);
hiprandState *prstate = &rstates[ithread];
for (i = ithread; i < nrows*ncols; i += nthreads) {
icol = i / nrows;
irow = i - icol * nrows;
umval = um[i];
sqrtss = ssq[i];
//    newsumsq = (nw * umval * umval) + (1 - nw) * sumsq;
sqrtnewss = hypotf(sqrtnw * umval, sqrt1mnw * sqrtss);
ssq[i] = sqrtnewss;
if (doupdate) {
mmval = mm[i];
veval = (nve > 1) ? ve[irow] : ve[0];
tsval = (nts > 1) ? ts[irow] : ts[0];
lrval = (nlr > 1) ? lr[irow] : lr[0];
sqrtnewss = hypotf(sqrtnewss, sqrteps);
denom = (veval == 0.5f) ? sqrtnewss : powf(sqrtnewss, veval*2);
grad = (umval / denom);
if (langevin > 0) grad += hiprand_normal(prstate) * langevin;
mmval += grad * lrval * tsval;
if (maskr > 0) {
if (maskr > 1) {
mmval *= mask[i];
} else {
mmval *= mask[icol];
}
}
mm[i] = mmval;
}
}
}