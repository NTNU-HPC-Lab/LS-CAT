#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __ADAGradn(int nrows, int ncols, float *mm, float *um, float *ssq, float *momentum, float mu, float *mask, int maskr, float nw, float *ve, int nve, float *ts, int nts, float *lr, int nlr, float langevin, float eps, int doupdate, hiprandState *rstates) {
int ithread = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int nthreads = blockDim.x * gridDim.x * gridDim.y;
int i, irow, icol;
float mmval, umval, sqrtss, sqrtnewss, veval, tsval, lrval, denom, grad, oldmom, newmom;
float sqrtnw = sqrtf(nw);
float sqrt1mnw = sqrtf(1-nw);
float sqrteps = sqrt(eps);
hiprandState *prstate = &rstates[ithread];
for (i = ithread; i < nrows*ncols; i += nthreads) {
icol = i / nrows;
irow = i - icol * nrows;
umval = um[i];
sqrtss = ssq[i];
//    newss = (nw * umval * umval) + (1 - nw) * sqval;
sqrtnewss = hypotf(sqrtnw * umval, sqrt1mnw * sqrtss);
ssq[i] = sqrtnewss;
if (doupdate) {
mmval = mm[i];
veval = (nve > 1) ? ve[irow] : ve[0];
tsval = (nts > 1) ? ts[irow] : ts[0];
lrval = (nlr > 1) ? lr[irow] : lr[0];
sqrtnewss = hypotf(sqrtnewss, sqrteps);
denom = (veval == 0.5f) ? sqrtnewss : powf(sqrtnewss, veval*2);
grad = (umval / denom);
if (langevin > 0) grad += hiprand_normal(prstate) * langevin;
grad = grad * lrval * tsval;               // Normal gradient
oldmom = momentum[i];                      // Momentum
newmom = grad + mu * oldmom;               // Compute new momentum
momentum[i] = newmom;                      // Save new momentum
mmval += newmom + mu * (newmom - oldmom);  // x_t = x_t-1 + p_t + mu(p_t - p_t-1)
if (maskr > 0) {
if (maskr > 1) {
mmval *= mask[i];
} else {
mmval *= mask[icol];
}
}
mm[i] = mmval;
}
}
}