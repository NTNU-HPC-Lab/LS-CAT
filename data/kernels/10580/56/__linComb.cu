#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __linComb(float *X, float wx, float *Y, float wy, float *Z, int len) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < len; i += blockDim.x * gridDim.x * gridDim.y) {
Z[i] = X[i]*wx + Y[i]*wy;
}
}