#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Programma che simula il comportamento del gpdt per
* la risoluzione di un kernel di una serie di
* valori di dimensione variabile utilizzando la
* tecnologia cuda.
* compilare con:
* nvcc -o simil_gpdt_si_cuda simil_gpdt_si_cuda.cu
* lanciare con:
* ./simil_gpdt_si_cuda [numero vettori] [numero componenti] [numero di righe da calcolare] [tipo di kernel] [grado(int)/sigma(float)]
**/

using namespace std;

/**
* Funzione che riempie i vettori con numeri
* casuali compresi tra 0 e 99.
**/
__global__ void Kernel_lineare(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

int j;
int pos;
int tmp_ind;
float lin;

for ( ; x < N ; x+=blockDim.x * gridDim.x)
{
for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
{
tmp_ind = ind[y];

lin = 0.0;

int Nr_val = Vnp[x];

for(j = 0; j < Nr_val; j++)
{
pos = Vp[x * nr_max_val + j];
lin =  lin + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
}

//Ris[x * dim_indici + y] = lin;
Ris[y * N + x ] = lin;
}
}
}