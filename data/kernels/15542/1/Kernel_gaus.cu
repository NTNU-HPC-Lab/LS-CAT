#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Programma che simula il comportamento del gpdt per
* la risoluzione di un kernel di una serie di
* valori di dimensione variabile utilizzando la
* tecnologia cuda.
* compilare con:
* nvcc -o simil_gpdt_si_cuda simil_gpdt_si_cuda.cu
* lanciare con:
* ./simil_gpdt_si_cuda [numero vettori] [numero componenti] [numero di righe da calcolare] [tipo di kernel] [grado(int)/sigma(float)]
**/

using namespace std;

/**
* Funzione che riempie i vettori con numeri
* casuali compresi tra 0 e 99.
**/
__global__ void Kernel_gaus(float *Vd, float *Ris, float *Nd, int N, int C, int dim_indici, int *ind, float sigma, int *Vp, int *Vnp, int nr_max_val)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

int j;
int pos;
int tmp_ind;
float gaus;

for ( ; x < N ; x+=blockDim.x * gridDim.x)
{
for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
{
tmp_ind = ind[y];

gaus = 0.0;

int Nr_val = Vnp[x];

for(j = 0; j < Nr_val; j++)
{
pos = Vp[x * nr_max_val + j];
gaus = gaus + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
}

gaus = - 2.0*gaus +Nd[x] + Nd[tmp_ind];
gaus = (exp(-gaus*sigma));

//Ris[x * dim_indici + y] = gaus;
Ris[y * N + x] = gaus;
}
}
}