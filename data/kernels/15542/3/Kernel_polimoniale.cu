#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Programma che simula il comportamento del gpdt per
* la risoluzione di un kernel di una serie di
* valori di dimensione variabile utilizzando la
* tecnologia cuda.
* compilare con:
* nvcc -o simil_gpdt_si_cuda simil_gpdt_si_cuda.cu
* lanciare con:
* ./simil_gpdt_si_cuda [numero vettori] [numero componenti] [numero di righe da calcolare] [tipo di kernel] [grado(int)/sigma(float)]
**/

using namespace std;

/**
* Funzione che riempie i vettori con numeri
* casuali compresi tra 0 e 99.
**/
__global__ void Kernel_polimoniale(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val, int s)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

int j;
int pos;
int tmp_ind;
float pol;
float tmp;

for ( ; x < N ; x+=blockDim.x * gridDim.x)
{
for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
{
tmp_ind = ind[y];

tmp = 1.0;

pol = 0.0;

int Nr_val = Vnp[x];

for(j = 0; j < Nr_val; j++)
{
pos = Vp[x * nr_max_val + j];
pol = pol + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
}

pol = pol + 1;

for(j = 0; j < s; j++)
{
tmp = tmp * pol;
}

//Ris[x * dim_indici + y] = tmp;
Ris[y * N + x ] = tmp;
}
}
}