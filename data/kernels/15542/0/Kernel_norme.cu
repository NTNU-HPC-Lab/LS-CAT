#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Programma che simula il comportamento del gpdt per
* la risoluzione di un kernel di una serie di
* valori di dimensione variabile utilizzando la
* tecnologia cuda.
* compilare con:
* nvcc -o simil_gpdt_si_cuda simil_gpdt_si_cuda.cu
* lanciare con:
* ./simil_gpdt_si_cuda [numero vettori] [numero componenti] [numero di righe da calcolare] [tipo di kernel] [grado(int)/sigma(float)]
**/

using namespace std;

/**
* Funzione che riempie i vettori con numeri
* casuali compresi tra 0 e 99.
**/
__global__ void Kernel_norme(float *Vd, float *Nd, int *Vp, int *Vnp, int N, int C, int nr_max_val)
{
long int x = threadIdx.x + blockIdx.x * blockDim.x;

int pos;

if(x < N)
{
float norma = 0;

int Nr_val = Vnp[x];

for(int i = 0; i < Nr_val; i++)
{
pos = Vp[x * nr_max_val + i];
norma = norma + (Vd[x * C + pos] * Vd[x * C + pos]);
}

Nd[x] = norma;
}

}