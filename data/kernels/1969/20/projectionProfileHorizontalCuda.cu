#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void projectionProfileHorizontalCuda( const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
const uint8_t * imageX = image + y * rowSize + x;
atomicAdd( &projection[x], (*imageX) );
}
}