#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setPixelCuda( uint8_t * in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * pointX, uint32_t * pointY, uint32_t pointSize, uint32_t value )
{
const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

if ( idPoint < pointSize) {
const uint32_t x = pointX[idPoint];
const uint32_t y = pointY[idPoint];
if ( x < width && y < height ) {
in[y * rowSize + x] = value;
}
}
}