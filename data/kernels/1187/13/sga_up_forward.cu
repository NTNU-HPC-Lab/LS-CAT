#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_up_forward (const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= n)
{
return;
}
int step = height * width;
//   int wsize=radius+1;

int base = index / width * step * depth + index % width;	//up->down
int fbase = index / width * step * wsize + index % width;

for (int row = height - 1; row >= 0; row--)
{
int shift = fbase + row * width;
for (int d = 0; d < depth; d++)
{
float temp = 0;
int location = base + d * step + row * width;
temp += top_data[location] * filters[shift];
if (row + 1 < height)
temp += top_data[location + width] * filters[shift + step];
else
temp += top_data[location] * filters[shift + step];
if (row + 2 < height)
temp +=
top_data[location + 2 * width] * filters[shift + 2 * step];
else
temp += top_data[location] * filters[shift + 2 * step];
if (row + 1 < height && d - 1 >= 0)
temp +=
top_data[location + width - step] * filters[shift + 3 * step];
else
temp += top_data[location] * filters[shift + 3 * step];
if (row + 1 < height && d + 1 < depth)
temp +=
top_data[location + width + step] * filters[shift + 4 * step];
else
temp += top_data[location] * filters[shift + 4 * step];

top_data[location] = temp;

}
}
}