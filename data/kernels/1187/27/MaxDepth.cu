#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MaxDepth (const int n, const float *bottom_data, const int step, const int depth, float *idx){

int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
int base = index / step * step * depth + index % step;
int k = 0;
for (int i = 1; i < depth; i++)
if (bottom_data[base + k * step] < bottom_data[base + i * step])
k = i;
idx[index] = k;
}