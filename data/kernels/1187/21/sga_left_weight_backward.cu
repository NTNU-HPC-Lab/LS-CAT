#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_left_weight_backward (const int n, const float *bottom_data, const float *top_data, const float *temp_diff, const int height, const int width, const int depth, const int wsize, float *filters_diff){

int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
int step = height * width;
int base = index / step * step * depth + index % step;	//up->down
int fbase = index / step * step * wsize + index % step;

//   int row = index%step/width;
int col = index % step % width;
for (int i = 0; i < depth; i++)
filters_diff[fbase] +=
temp_diff[base + i * step] * bottom_data[base + i * step];
if (col + 1 < width)
{
int location = fbase + step;
for (int i = 0; i < depth; i++)
filters_diff[location] +=
temp_diff[base + i * step] * top_data[base + i * step + 1];

location = fbase + 3 * step;
filters_diff[location] += temp_diff[base] * bottom_data[base];
for (int i = 1; i < depth; i++)
filters_diff[location] +=
temp_diff[base + i * step] * top_data[base + (i - 1) * step + 1];

location = fbase + 4 * step;
filters_diff[location] +=
temp_diff[base + (depth - 1) * step] * bottom_data[base + (depth - 1) * step];
for (int i = 0; i < depth - 1; i++)
filters_diff[location] +=
temp_diff[base + i * step] * top_data[base + (i + 1) * step + 1];
}
/*
else{
//int location = fbase + step;
for(int i=0; i<depth; i++){
float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
}*/
if (col + 2 < width)
{
int location = fbase + 2 * step;
for (int i = 0; i < depth; i++)
filters_diff[location] +=
temp_diff[base + i * step] * top_data[base + i * step + 2];
}
/*
else{
int location = fbase + 2*step;
for(int i=0; i<depth; i++)
filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
}
*/
}