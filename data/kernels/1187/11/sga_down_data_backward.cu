#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_down_data_backward (const int n, const float *filters, float *top_diff, const int height, const int width, const int depth, const int wsize, float *bottom_diff){
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
int step = height * width;
int base = index / width * step * depth + index % width;	//up->down
int fbase = index / width * step * wsize + index % width;
for (int row = height - 1; row >= 0; row--)
{
int shift = fbase + row * width;
for (int d = 0; d < depth; d++)
{
int location = base + d * step + row * width;
float temp = top_diff[location];
if (row + 1 < height)
temp +=
top_diff[location + width] * filters[shift + width + step];
if (row + 2 < height)
temp +=
top_diff[location + 2 * width] * filters[shift + 2 * width +
2 * step];
if (row + 1 < height && d + 1 < depth)
temp +=
top_diff[location + width + step] * filters[shift + width +
3 * step];
if (row + 1 < height && d - 1 >= 0)
temp +=
top_diff[location + width - step] * filters[shift + width +
4 * step];
top_diff[location] = temp;
bottom_diff[location] += temp * filters[shift];
}
}
/*	for(int d = 0; d < depth; d ++){
int shift = fbase;
int location = base + d * step;
bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]);
//       bottom_diff[location] += top_diff[location];
shift += width;
location += width;
bottom_diff[location] += top_diff[location] * filters[shift + 2*step];
}
for(int row=1;row<height;row++){
int location = base + row * width;
int shift = fbase + row * width;
bottom_diff[location] += top_diff[location] * filters[shift + 3*step];
location += (depth - 1)*step;
bottom_diff[location] += top_diff[location] * filters[shift + 4*step];
}
*/
for (int row = 0; row < height; row++)
{
int location = base + row * width;
int shift = fbase + row * width;
bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
location += (depth - 1) * step;
bottom_diff[location] += top_diff[location] * filters[shift + 4 * step];
}
}