#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lga_data_backward (const int n, const float *filters, const float *top_diff, const int height, const int width, const int channel, const int radius, float *bottom_diff){
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= n)
{
return;
}
int step = height * width;
int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
int fbase =
index / (step * channel) * (step * wsize * wsize * 3) + index % step;
int row = index % step / width;
int col = index % width;
int depth = index / step % channel;
for (int d = -1; d <= 1; d++)
{
for (int r = -radius; r <= radius; r++)
{
for (int c = -radius; c <= radius; c++)
{
int rr = r + row;
int cc = c + col;
int dd = d + depth;
//      int shift = 0;
if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
&& dd < channel)
{
int shift = r * width + c + d * step;
//      int fshift= r*width+c;
int location =
(-d + 1) * (wsize * wsize) + (-r + radius) * wsize - c +
radius;
bottom_diff[index] +=
top_diff[index + shift] * filters[fbase + r * width + c +
location * step];
}
else
{
int location =
(d + 1) * (wsize * wsize) + (r + radius) * wsize + c +
radius;
bottom_diff[index] +=
top_diff[index] * filters[fbase + location * step];
}
}
}
}
}