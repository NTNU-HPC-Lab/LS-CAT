#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_down_forward (const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data){
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
int step = height * width;
//  int wsize=radius+1;
int base = index / width * step * depth + index % width;	//up->down
int fbase = index / width * step * wsize + index % width;


for (int row = 0; row < height; row++)
{
int shift = fbase + row * width;
for (int d = 0; d < depth; d++)
{
float temp = 0;
int location = base + d * step + row * width;
temp += top_data[location] * filters[shift];
if (row - 1 >= 0)
temp += top_data[location - width] * filters[shift + step];
else
temp += top_data[location] * filters[shift + step];
if (row - 2 >= 0)
temp +=
top_data[location - 2 * width] * filters[shift + 2 * step];
else
temp += top_data[location] * filters[shift + 2 * step];
if (row - 1 >= 0 && d - 1 >= 0)
temp +=
top_data[location - width - step] * filters[shift + 3 * step];
else
temp += top_data[location] * filters[shift + 3 * step];
if (row - 1 >= 0 && d + 1 < depth)
temp +=
top_data[location - width + step] * filters[shift + 4 * step];
else
temp += top_data[location] * filters[shift + 4 * step];
//                      if(top_data[locaiton]<temp)
//                      mask[location]=0;
top_data[location] = temp;

}
}
}