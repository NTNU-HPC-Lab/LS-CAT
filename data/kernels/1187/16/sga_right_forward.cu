#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_right_forward (const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= n)
{
return;
}
int step = height * width;
//   int wsize=radius+1;

int base = index / height * step * depth + (index % height) * width;	//up->down
int fbase = index / height * step * wsize + (index % height) * width;

for (int col = 0; col < width; col++)
{
int shift = fbase + col;
for (int d = 0; d < depth; d++)
{
float temp = 0;
int location = base + d * step + col;
temp += top_data[location] * filters[shift];
if (col - 1 >= 0)
temp += top_data[location - 1] * filters[shift + step];
else
temp += top_data[location] * filters[shift + step];
if (col - 2 >= 0)
temp += top_data[location - 2] * filters[shift + 2 * step];
else
temp += top_data[location] * filters[shift + 2 * step];
if (col - 1 >= 0 && d - 1 >= 0)
temp += top_data[location - 1 - step] * filters[shift + 3 * step];
else
temp += top_data[location] * filters[shift + 3 * step];
if (col - 1 >= 0 && d + 1 < depth)
temp += top_data[location - 1 + step] * filters[shift + 4 * step];
else
temp += top_data[location] * filters[shift + 4 * step];

top_data[location] = temp;
}
}
}