#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Max (const int n, const float *top_temp, float *top_data, float *mask, const int mask_index){
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
if (top_data[index] < top_temp[index])
{
top_data[index] = top_temp[index];
mask[index] = mask_index;
}
}