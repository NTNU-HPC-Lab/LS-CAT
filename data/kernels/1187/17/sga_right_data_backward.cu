#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_right_data_backward (const int n, const float *filters, float *top_diff, const int height, const int width, const int depth, const int wsize, float *bottom_diff){
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
int step = height * width;
int base = index / height * step * depth + (index % height) * width;	//up->down
int fbase = index / height * step * wsize + (index % height) * width;

for (int col = width - 1; col >= 0; col--)
{
int shift = fbase + col;
for (int d = 0; d < depth; d++)
{
int location = base + d * step + col;
float temp = top_diff[location];
if (col + 1 < width)
temp += top_diff[location + 1] * filters[shift + 1 + step];
if (col + 2 < width)
temp += top_diff[location + 2] * filters[shift + 2 + 2 * step];
if (col + 1 < width && d + 1 < depth)
temp +=
top_diff[location + 1 + step] * filters[shift + 1 + 3 * step];
if (col + 1 < width && d - 1 >= 0)
temp +=
top_diff[location + 1 - step] * filters[shift + 1 + 4 * step];
top_diff[location] = temp;
bottom_diff[location] += (temp * filters[shift]);
}
}
/*
for(int d = 0; d < depth; d ++){
int shift = fbase;// + width*(height-1);
int location = base;// + width*(height-1) + d * step;
bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]);
//       bottom_diff[location] += top_diff[location];
shift += 1;
location += 1;
bottom_diff[location] += top_diff[location] * filters[shift + 2*step];
}
for(int col=1;col<width;col++){
int shift = fbase + col;
int location = base + col;
bottom_diff[location] += top_diff[location] * filters[shift + 3*step];
location += (depth - 1)*step;
bottom_diff[location] += top_diff[location] * filters[shift + 4*step];
}*/
for (int col = 0; col < width; col++)
{
int shift = fbase + col;
int location = base + col;
bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
location += (depth - 1) * step;
bottom_diff[location] += top_diff[location] * filters[shift + 4 * step];
}
}