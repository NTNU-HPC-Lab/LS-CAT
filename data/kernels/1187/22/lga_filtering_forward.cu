#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lga_filtering_forward (const int n, const float *bottom_data, const float *filters, const int height, const int width, const int channel, const int radius, float *top_data){
int index = blockIdx.x * blockDim.x + threadIdx.x;
//    printf("OK\n");
//    printf("%d, %.2f, %.2f\n", index, bottom_data[index], top_data[index]);
if (index >= n)
{
return;
}
//    top_data[index]=1.0;
//    assert(0);
int step = height * width;
int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
int fbase =
index / (step * channel) * (step * wsize * wsize * 3) + index % step;
int row = index % step / width;
int col = index % width;
int depth = index / step % channel;
for (int d = -1; d <= 1; d++)
{
for (int r = -radius; r <= radius; r++)
{
for (int c = -radius; c <= radius; c++)
{
int rr = r + row;
int cc = c + col;
int dd = d + depth;
int shift = 0;
if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
&& dd < channel)
shift = r * width + c + d * step;
int location =
(d + 1) * (wsize * wsize) + (r + radius) * wsize + c + radius;
top_data[index] +=
bottom_data[index + shift] * filters[fbase + location * step];
}
}
}
//        top_data[index]=1.0;
//        printf("%d, %d, %d, %.2f, %.2f\n", index, row, col, bottom_data[index], top_data[index]);
}