#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_temp_grad (const int n, const float *gradOutput, const float *mask, float *top_grad, const int mask_index){
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= n)
{
return;
}
if (((int) mask[index]) == mask_index)
top_grad[index] = gradOutput[index];
}