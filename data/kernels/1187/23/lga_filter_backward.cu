#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lga_filter_backward (const int n, const float *bottom_data, const float *top_diff, const int height, const int width, const int channel, const int radius, float *filter_diff){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= n)
{
return;
}
int step = height * width;
int wsize = 2 * radius + 1;

int base =
index / (step * wsize * wsize * 3) * (step * channel) + index % step;
int location = index / step % (wsize * wsize * 3);
int d = location / (wsize * wsize) - 1;
int r = (location / wsize) % wsize - radius;
int c = location % wsize - radius;

int rr = index % step / width + r;
int cc = index % width + c;

for (int i = 0; i < channel; i++)
{
int dd = i + d;
if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
&& dd < channel)
{
int shift = r * width + c + d * step;
filter_diff[index] +=
top_diff[base + i * step] * bottom_data[base + shift + i * step];
}
else
filter_diff[index] +=
top_diff[base + i * step] * bottom_data[base + i * step];
}



}