#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void recipSummation(double* data, double* recip, int len)
{
const int y = blockIdx.y * gridDim.x * blockDim.x;
const int x = blockIdx.x * blockDim.x;
const int i = threadIdx.x + x + y;
if (i < len) {
const int j = 2 * i;
data[j]     *= recip[i];
data[j + 1] *= recip[i];
}
}