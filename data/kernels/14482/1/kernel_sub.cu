#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
int tmp = 0;
int i = threadIdx.x;
#if __CUDA_ARCH__>=200
//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
if (i == 0) return;

//for (int i = size_biggest - 1; i >= 0; i--) {
if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
tmp = first[i - 1] - second[i-1-diff];
} else if (first[i - 1] != '+' && first[i - 1] != '-') {
tmp = first[i - 1];
}

if (tmp < 0) {
// warning 10 - tmp ?
newB[i - 1]--;
tmp += 10;
}
if (i != 0)
newB[i] += tmp;
//}
}