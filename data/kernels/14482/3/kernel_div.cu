#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_div(char* newB, char* first, char* second, int size_first, int size_second, int * size_newB, char* aux) {
int i = threadIdx.x;
int j = threadIdx.y;

if(j==0 && i==0){
if(first[j]=='-' || second[i]=='-')
newB[0]='-';
else
newB[0]='+';
return;
}

#if __CUDA_ARCH__>=200
printf("#i, j = %d, %d\n", i, j);
#endif
// adapted from kernel_sub
int diff = size_first - size_second;
int tmp = 0;
if (j - 1 - diff >= 0 && (second[j - 1 - diff] != '+' && second[j - 1 - diff] != '-')) {
tmp = first[j - 1] - second[j-1-diff];
} else if (first[j - 1] != '+' && first[j - 1] != '-') {
tmp = first[j - 1];
}

if (tmp < 0) {
// warning 10 - tmp ?
aux[i * size_first + j - 1]--;
tmp += 10;
}
if (i != 0)
aux[i * size_first + j] += tmp;
// end of kernel_sub

#if __CUDA_ARCH__>=200
printf("#aux = %d\n", aux[i * size_first + j]);
#endif

/*
char* temp = NULL;
//init(size_second + 1, temp);
int t = 0; // temp's index
int n = 0; // newB's index
for (int i = size_first - 1; i >= 0; i -= t) {
t = 0;
for (int j = i - size_second; j <= i; j++) {
if (j >= 0) {
temp[t] = first[j];
t++;
}
}
// verify that we are not attempting to divide something too small
if (isFirstBiggerThanSecond(second, temp, size_second)) {
t = 0;
for (int j = i - size_second - 1; j <= i; j++) {
if (j < 0) {
// nothing left to divide, exit function
return;
} else {
temp[t] = first[j];
t++;
}
}
}
// now that we have our thing, let's get to the division itself
char res = 0;
char* sub_res = NULL;
int size_res = 0;
//init(size_second, sub_res);
do {
//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
res++;
} while (0); //sub_res > 0
// current division done, save result & move on to the next
newB[n] = res;
n++;
}
// all divisions done, we need to realign our result;
int diff = size_second - n;
for (int i = size_second - 1; i > n; i++) {
newB[i] = newB[i - diff];
}*/
}