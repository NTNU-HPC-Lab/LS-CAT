#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {

int i = threadIdx.x;
int j = threadIdx.y;

int tid = j * gridDim.x * blockDim.x + i ;

if(j!=0 && i!=0){
newB[tid] = first[i] * second[j];
}

if(j==0 && i==0){
if(first[j] != second[i])
newB[0]='-';
else
newB[0]='+';
}
}