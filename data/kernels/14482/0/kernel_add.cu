#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline void charAtomicAdd(char *address, char value) {
int oldval, newval, readback;

oldval = *address;
newval = oldval + value;
while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
oldval = readback;
newval = oldval + value;
}
}
__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
int tmp = 0;
int i = threadIdx.x;
#if __CUDA_ARCH__>=200
//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
if (i == 0) return;

//for (int i = size_biggest - 1; i >= 0; i--) {
if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
tmp = second[i - 1 - diff] + first[i - 1];
} else if (first[i - 1] != '+' && first[i - 1] != '-') {
tmp = first[i - 1];
}

if (tmp >= 10) {
//charAtomicAdd(&newB[i], 1);
newB[i - 1]++;
tmp = tmp % 10;
}
if (i != 0)
newB[i] += tmp;
//}
}