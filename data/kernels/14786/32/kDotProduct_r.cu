#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
__shared__ float shmem[DP_BLOCKSIZE];

uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
shmem[threadIdx.x] = 0;
if (eidx < numCols) {
for (; eidx < numElements; eidx += numCols) {
shmem[threadIdx.x] += a[eidx] * b[eidx];
}
}
__syncthreads();
if (threadIdx.x < 256) {
shmem[threadIdx.x] += shmem[threadIdx.x + 256];
}
__syncthreads();
if (threadIdx.x < 128) {
shmem[threadIdx.x] += shmem[threadIdx.x + 128];
}
__syncthreads();
if (threadIdx.x < 64) {
shmem[threadIdx.x] += shmem[threadIdx.x + 64];
}
__syncthreads();
if (threadIdx.x < 32) {
volatile float* mysh = &shmem[threadIdx.x];
*mysh += mysh[32];
*mysh += mysh[16];
*mysh += mysh[8];
*mysh += mysh[4];
*mysh += mysh[2];
*mysh += mysh[1];
if (threadIdx.x == 0) {
target[blockIdx.x] = *mysh;
}
}
}