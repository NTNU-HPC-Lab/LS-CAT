#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpack_top( const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
const int x_inner = x - 2*halo_depth;

const int gid = threadIdx.x+blockDim.x*blockIdx.x;
if(gid >= x_inner*depth) return;

const int lines = gid / x_inner;
const int offset = x*(y - halo_depth) + lines*2*halo_depth;
field[offset+gid] = buffer[gid];
}