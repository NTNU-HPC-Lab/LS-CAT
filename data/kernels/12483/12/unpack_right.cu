#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpack_right( const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
const int y_inner = y - 2*halo_depth;

const int gid = threadIdx.x+blockDim.x*blockIdx.x;
if(gid >= y_inner*depth) return;

const int lines = gid / depth;
const int offset = x - halo_depth + lines*(x - depth);
field[offset+gid] = buffer[gid];
}