#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_chunk_data( int x, int y, double dx, double dy, double* cell_x, double* cell_y, double* cell_dx, double* cell_dy, double* vertex_x, double* vertex_y, double* volume, double* x_area, double* y_area)
{
const int gid = blockIdx.x*blockDim.x+threadIdx.x;

if(gid < x)
{
cell_x[gid] = 0.5*(vertex_x[gid]+vertex_x[gid+1]);
cell_dx[gid] = dx;
}

if(gid < y)
{
cell_y[gid] = 0.5*(vertex_y[gid]+vertex_y[gid+1]);
cell_dy[gid] = dy;
}

if(gid < x*y)
{
volume[gid] = dx*dy;
}

if(gid < (x+1)*y)
{
x_area[gid] = dy;
}

if(gid < x*(y+1))
{
y_area[gid] = dx;
}
}