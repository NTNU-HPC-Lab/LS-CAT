#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "set_chunk_data.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int x = 1;
int y = 1;
double dx = 1;
double dy = 1;
double *cell_x = NULL;
hipMalloc(&cell_x, XSIZE*YSIZE);
double *cell_y = NULL;
hipMalloc(&cell_y, XSIZE*YSIZE);
double *cell_dx = NULL;
hipMalloc(&cell_dx, XSIZE*YSIZE);
double *cell_dy = NULL;
hipMalloc(&cell_dy, XSIZE*YSIZE);
double *vertex_x = NULL;
hipMalloc(&vertex_x, XSIZE*YSIZE);
double *vertex_y = NULL;
hipMalloc(&vertex_y, XSIZE*YSIZE);
double *volume = NULL;
hipMalloc(&volume, XSIZE*YSIZE);
double *x_area = NULL;
hipMalloc(&x_area, XSIZE*YSIZE);
double *y_area = NULL;
hipMalloc(&y_area, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
set_chunk_data<<<gridBlock,threadBlock>>>(x,y,dx,dy,cell_x,cell_y,cell_dx,cell_dy,vertex_x,vertex_y,volume,x_area,y_area);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
set_chunk_data<<<gridBlock,threadBlock>>>(x,y,dx,dy,cell_x,cell_y,cell_dx,cell_dy,vertex_x,vertex_y,volume,x_area,y_area);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
set_chunk_data<<<gridBlock,threadBlock>>>(x,y,dx,dy,cell_x,cell_y,cell_dx,cell_dy,vertex_x,vertex_y,volume,x_area,y_area);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}