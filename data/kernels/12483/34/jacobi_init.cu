#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobi_init( const int x_inner, const int y_inner, const int halo_depth, const double* density, const double* energy, const double rx, const double ry, double* kx, double* ky, double* u0, double* u, const int coefficient)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

const double u_temp = energy[index]*density[index];
u0[index] = u_temp;
u[index] = u_temp;

if(row == 0 || col == 0) return;

double density_center;
double density_left;
double density_down;

if(coefficient == CONDUCTIVITY)
{
density_center = density[index];
density_left = density[index-1];
density_down = density[index-x];
}
else if(coefficient == RECIP_CONDUCTIVITY)
{
density_center = 1.0/density[index];
density_left = 1.0/density[index-1];
density_down = 1.0/density[index-x];
}

kx[index] = rx*(density_left+density_center) /
(2.0*density_left*density_center);
ky[index] = ry*(density_down+density_center) /
(2.0*density_down*density_center);
}