#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "field_summary.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int x_inner = 1;
const int y_inner = 1;
const int halo_depth = 1;
const double *volume = NULL;
hipMalloc(&volume, XSIZE*YSIZE);
const double *density = NULL;
hipMalloc(&density, XSIZE*YSIZE);
const double *energy0 = NULL;
hipMalloc(&energy0, XSIZE*YSIZE);
const double *u = NULL;
hipMalloc(&u, XSIZE*YSIZE);
double *vol_out = NULL;
hipMalloc(&vol_out, XSIZE*YSIZE);
double *mass_out = NULL;
hipMalloc(&mass_out, XSIZE*YSIZE);
double *ie_out = NULL;
hipMalloc(&ie_out, XSIZE*YSIZE);
double *temp_out = NULL;
hipMalloc(&temp_out, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
field_summary<<<gridBlock,threadBlock>>>(x_inner,y_inner,halo_depth,volume,density,energy0,u,vol_out,mass_out,ie_out,temp_out);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
field_summary<<<gridBlock,threadBlock>>>(x_inner,y_inner,halo_depth,volume,density,energy0,u,vol_out,mass_out,ie_out,temp_out);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
field_summary<<<gridBlock,threadBlock>>>(x_inner,y_inner,halo_depth,volume,density,energy0,u,vol_out,mass_out,ie_out,temp_out);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}