#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zero_buffer( const int x, const int y, double* buffer)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;

if(gid < x*y)
{
buffer[gid] = 0.0;
}
}