#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobi_copy_u( const int x_inner, const int y_inner, const double* src, double* dest)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;

if(gid < x_inner*y_inner)
{
dest[gid] = src[gid];
}
}