#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ppcg_init( const int x_inner, const int y_inner, const int halo_depth, const double theta, const double* r, double* sd)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

sd[index] = r[index] / theta;
}