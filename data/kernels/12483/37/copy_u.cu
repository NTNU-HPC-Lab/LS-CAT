#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_u( const int x_inner, const int y_inner, const int halo_depth, const double* src, double* dest)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

dest[index] = src[index];
}