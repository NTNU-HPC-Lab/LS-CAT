#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void finalise( const int x_inner, const int y_inner, const int halo_depth, const double* density, const double* u, double* energy)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

energy[index] = u[index]/density[index];
}