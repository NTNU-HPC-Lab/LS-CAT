#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cg_calc_p( const int x_inner, const int y_inner, const int halo_depth, const double beta, const double* r, double* p)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

p[index] = r[index] + beta*p[index];
}