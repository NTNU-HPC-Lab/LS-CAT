#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "set_chunk_data_vertices.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int x = 1;
int y = 1;
int halo_depth = 1;
double dx = 1;
double dy = 1;
double x_min = 1;
double y_min = 1;
double *vertex_x = NULL;
hipMalloc(&vertex_x, XSIZE*YSIZE);
double *vertex_y = NULL;
hipMalloc(&vertex_y, XSIZE*YSIZE);
double *vertex_dx = NULL;
hipMalloc(&vertex_dx, XSIZE*YSIZE);
double *vertex_dy = NULL;
hipMalloc(&vertex_dy, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
set_chunk_data_vertices<<<gridBlock,threadBlock>>>(x,y,halo_depth,dx,dy,x_min,y_min,vertex_x,vertex_y,vertex_dx,vertex_dy);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
set_chunk_data_vertices<<<gridBlock,threadBlock>>>(x,y,halo_depth,dx,dy,x_min,y_min,vertex_x,vertex_y,vertex_dx,vertex_dy);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
set_chunk_data_vertices<<<gridBlock,threadBlock>>>(x,y,halo_depth,dx,dy,x_min,y_min,vertex_x,vertex_y,vertex_dx,vertex_dy);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}