#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_chunk_data_vertices( int x, int y, int halo_depth, double dx, double dy, double x_min, double y_min, double* vertex_x, double* vertex_y, double* vertex_dx, double* vertex_dy)
{
const int gid = blockIdx.x*blockDim.x+threadIdx.x;

if(gid < x+1)
{
vertex_x[gid] = x_min + dx*(gid-halo_depth);
vertex_dx[gid] = dx;
}

if(gid < y+1)
{
vertex_y[gid] = y_min + dy*(gid-halo_depth);
vertex_dy[gid] = dy;
}
}