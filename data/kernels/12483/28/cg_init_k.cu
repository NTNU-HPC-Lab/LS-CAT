#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cg_init_k( const int x_inner, const int y_inner, const int halo_depth, const double* w, double* kx, double* ky, double rx, double ry)
{
const int gid = threadIdx.x+blockIdx.x*blockDim.x;
if(gid >= x_inner*y_inner) return;

const int x = x_inner + 2*halo_depth-1;
const int col = gid % x_inner;
const int row = gid / x_inner;
const int off0 = halo_depth*(x + 1);
const int index = off0 + col + row*x;

kx[index] = rx*(w[index-1]+w[index]) /
(2.0*w[index-1]*w[index]);
ky[index] = ry*(w[index-x]+w[index]) /
(2.0*w[index-x]*w[index]);
}