#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
size_t index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= len)
return;

d_grad[index] = 0.0f;

for (size_t j = 0; j < len; ++j)
{
d_grad[index] += A[index * len + j] * d_x[j];
}

d_grad[index] *= 2.0f;
d_grad[index] += b[index];
}