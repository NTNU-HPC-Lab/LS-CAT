#include "hip/hip_runtime.h"
#include "includes.h"


// Calculate d[n] = a[n]*b[n] + c[n]

__global__ void custom_kernel(float *a, float *b, float *c, float *d, int N) {
int idx = blockDim.x*blockIdx.x + threadIdx.x;
int num_threads = blockDim.x * gridDim.x;
while(idx < N) {
d[idx] = a[idx]*b[idx]+c[idx];
idx += num_threads;
}
}