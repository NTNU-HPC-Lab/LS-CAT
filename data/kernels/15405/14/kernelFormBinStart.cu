#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFormBinStart (	int* devOutputBinStart, unsigned int* devInputBinCirPairBin, unsigned int  bcPairLen)
{

__shared__ int cache[257]; //256 bcpair + the last bc pair in the previous block

int bcPairIdx = blockDim.x * blockIdx.x + threadIdx.x;

if (bcPairIdx >= bcPairLen)
{
return;
}

cache[1 + threadIdx.x] = devInputBinCirPairBin[bcPairIdx];

if ( threadIdx.x == 0 )
{
if ( bcPairIdx != 0 )
{
cache[0] = devInputBinCirPairBin[bcPairIdx - 1];
}
else
{
cache[0] = -1;
}
}

__syncthreads();

if (cache[1 + threadIdx.x] != cache[threadIdx.x])
{
//printf("b: %d, s: %d\n", cache[1 + threadIdx.x], bcPairIdx);
devOutputBinStart[cache[1 + threadIdx.x]] = bcPairIdx;
}
}