#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addInc(unsigned int* deviceInput, unsigned int* deviceOutput, int eleCnt, unsigned int* deviceInc)
{
/*
__shared__ int inc;
if (threadIdx.x == 0)
{
inc = deviceInc[blockIdx.x];
}
__syncthreads();
*/
int inc = deviceInc[blockIdx.x];

int cntInB = blockDim.x * 2;
int idxInG = blockIdx.x * cntInB + threadIdx.x;

if (idxInG < eleCnt)
{
deviceOutput[idxInG] = deviceInput[idxInG] + inc;
}

if (idxInG + blockDim.x < eleCnt)
{
deviceOutput[idxInG + blockDim.x] = deviceInput[idxInG + blockDim.x] + inc;
}

}