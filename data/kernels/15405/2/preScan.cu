#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void preScan(unsigned int* deviceInput, unsigned int* deviceOutput, int cnt, unsigned int* deviceSum)
{
extern __shared__ unsigned int temp[];
int cntInB = blockDim.x * 2;
int idxInG = cntInB * blockIdx.x + threadIdx.x;

int idxInB = threadIdx.x;
temp[2 * idxInB]		= 0;
temp[2 * idxInB +1]		= 0;

if (idxInG < cnt)
{
temp[idxInB] = deviceInput[idxInG];
}

if (idxInG + blockDim.x < cnt)
{
temp[idxInB + blockDim.x] = deviceInput[idxInG + blockDim.x];
}

int offset = 1;
for (int d = cntInB >> 1; d > 0; d>>=1)
{
__syncthreads();
if (threadIdx.x < d)
{
int ai = offset - 1 + offset * (threadIdx.x * 2);
int bi = ai + offset;
temp[bi] += temp[ai];
}
offset *= 2;
}

__syncthreads();
//before clear the last element, move the last element to deviceSums.
if (threadIdx.x == 0)
{
deviceSum[blockIdx.x] = temp[cntInB - 1];
temp[cntInB - 1] = 0;
}

//downsweep
for (int d = 1; d < cntInB; d *=2)
{
offset >>= 1;
__syncthreads();

if (threadIdx.x < d)
{

int ai = offset - 1 + offset * (threadIdx.x * 2);
int bi = ai + offset;
unsigned int be = temp[bi];
temp[bi] += temp[ai];
temp[ai] = be;
}
}

if (idxInG < cnt)
{
deviceOutput[idxInG] = temp[idxInB];
}

if (idxInG + blockDim.x < cnt)
{
deviceOutput[idxInG + blockDim.x] = temp[idxInB + blockDim.x];
}
}