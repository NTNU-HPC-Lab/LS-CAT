#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void toGrayScale(unsigned char *output, unsigned char *input, int width, int height, int components)
{
int column = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

if (row >= height || column >= width)
return;

int index = column + row * width;
unsigned char * threadData = input + components * index;
unsigned char * outputData = output + index;

const float partRed = 0.299f;
const float partGreen = 0.587f;
const float partBlue = 0.114;

unsigned char greyScale = partBlue * threadData[0] + partGreen * threadData[1] + partRed * threadData[2];

outputData[0] = greyScale;
}