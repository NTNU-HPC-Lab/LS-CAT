#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y >= height || x >= width)
return;

const int BLOCK_SIZE = 16;

// Where does our data start
int blockStartIndexX = blockIdx.x * blockDim.x - 1;
int blockStartIndexY = blockIdx.y * blockDim.y - 1;

// Clamp to edge
if (blockStartIndexX < 0)
blockStartIndexX = 0;

if (blockStartIndexX >= width)
blockStartIndexX = blockDim.x - 1;

if (blockStartIndexY < 0)
blockStartIndexY = 0;

if (blockStartIndexY >= height)
blockStartIndexY = blockDim.y - 1;

// Shared Data
__shared__ unsigned char pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

// Where is our data
unsigned char* cacheInput = input + (blockStartIndexX + blockStartIndexY * width);

// Linear index (16x16 -> 0..255)
int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;

int maxLoadSizeBytes = (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2); // 18x18 Block -> 324 Bytes
int maxIndexBytes = maxLoadSizeBytes / sizeof(short); // 18x18 Block -> Index 162

if (threadIndex < maxIndexBytes)
{
// Calculate offset
int offsetInBytes = threadIndex * sizeof(short);

int block_half = (BLOCK_SIZE + 2) / 2;
int byteRow = offsetInBytes / (BLOCK_SIZE + 2);
int byteCol = threadIndex % block_half * 2;

int offset = byteCol + byteRow * width;
//int offsetBuffer = byteCol + byteRow * (BLOCK_SIZE + 2);

// Copy Data
unsigned char* toLoad = cacheInput + offset;
/**(&pixels[0][0] + offsetBuffer) = *toLoad;
*(&pixels[0][0] + offsetBuffer + 1) = *(toLoad + 1);*/

pixels[byteRow][byteCol] = *toLoad;
pixels[byteRow][byteCol + 1] = *(toLoad + 1);
}

__syncthreads();


// Sobel weights
float weightsX[9] = { -1, -2, -1,
0,  0,  0,
1,  2,  1 };

float weightsY[9] = { -1,  0,  1,
-2,  0,  2,
-1,  0,  1 };

int offsetY[9] = { -1,  -1,  -1,
0,   0,   0,
1,   1,   1 };

int offsetX[9] = { -1,   0,   1,
-1,   0,   1,
-1,   0,   1 };


float pointX = 0.f;
float pointY = 0.f;

#pragma unroll
for (int i = 0; i < 9; i++)
{
int indexX = threadIdx.x + 1 + offsetX[i];
int indexY = threadIdx.y + 1 + offsetY[i];

unsigned char pixel = pixels[indexY][indexX];
pointX += pixel * weightsX[i];
pointY += pixel * weightsY[i];
}


// Do Sobel here!
int index = x + y * width;
unsigned char * outputData = output + index;
outputData[0] = sqrtf(pointX * pointX + pointY * pointY);
}