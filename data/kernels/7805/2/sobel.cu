#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y >= height || x >= width)
return;

// Sobel weights
float weightsX[9] = { -1, -2, -1,
0,  0,  0,
1,  2,  1 };

float weightsY[9] = { -1,  0,  1,
-2,  0,  2,
-1,  0,  1 };

int offsetY[9] = { -1,  -1,  -1,
0,   0,   0,
1,   1,   1 };

int offsetX[9] = { -1,   0,   1,
-1,   0,   1,
-1,   0,   1 };


float pointX = 0.f;
float pointY = 0.f;
#pragma unroll
for (int i = 0; i < 9; i++)
{
int index = (x + offsetX[i]) + (y + offsetY[i]) * width;

unsigned char pixel = *(input + index);
pointX += pixel * weightsX[i];
pointY += pixel * weightsY[i];
}


// Do Sobel here!
int index = x + y * width;
unsigned char * outputData = output + index;
outputData[0] = sqrtf(pointX * pointX + pointY * pointY);
}