#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void toOneChannel(unsigned char *data, int width, int height, int components)
{
int column = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

if (row >= height || column >= width)
return;

unsigned char * threadData = data + (components * (column + row * width));

for (int i = 0; i < components - 1; i++)
{
threadData[i] = 0;
}
}