#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogramm(float* hist, unsigned char* input, int width, int height, int stride)
{
int index = blockIdx.x * blockDim.x * stride + threadIdx.x;
int size = width * height;
if (index > size - 1)
return;

__shared__ unsigned int histo_private[256];

#pragma unroll
for (int i = 0; i < 8; i++)
{
histo_private[threadIdx.x * 8 + i] = 0;
}

__syncthreads();

int i = 0;
while (i < stride && index < size)
{
int pixel = input[index];
atomicAdd(&(histo_private[pixel]), 1);
index += blockDim.x;
i++;
}

__syncthreads();

#pragma unroll
for (int i = 0; i < 8; i++)
{
int x_off = threadIdx.x * 8 + i;
hist[x_off * 3 + 0] = (x_off - 128.f) / 256.f * (float)width;

float factor = .48f;
float scaledValue = ((float)(histo_private[x_off]) / (float)size) - (factor / gridDim.x);
atomicAdd(&(hist[x_off * 3 + 1]), scaledValue * (float)height);
}
}