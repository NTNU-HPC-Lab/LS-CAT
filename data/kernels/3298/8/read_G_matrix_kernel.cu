#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void read_G_matrix_kernel(int S, int vertex_index, int* i_index, int* j_index, bool* is_Bennett, double* exp_Vj, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* result_ptr, int incr) {
int l = threadIdx.x;

double result, delta;

if (j_index[l] < vertex_index) {
delta = i_index[l] == j_index[l] ? 1. : 0.;
result = (N_ptr[i_index[l] + LD_N * j_index[l]] * exp_Vj[l] - delta) / (exp_Vj[l] - 1.);
}
else
result = G_ptr[i_index[l] + LD_G * (j_index[l] - vertex_index)];

result_ptr[l * incr] = is_Bennett[l] ? 0. : result;
}