#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "read_G_matrix_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int S = 1;
int vertex_index = 1;
int *i_index = NULL;
hipMalloc(&i_index, XSIZE*YSIZE);
int *j_index = NULL;
hipMalloc(&j_index, XSIZE*YSIZE);
bool *is_Bennett = NULL;
hipMalloc(&is_Bennett, XSIZE*YSIZE);
double *exp_Vj = NULL;
hipMalloc(&exp_Vj, XSIZE*YSIZE);
double *N_ptr = NULL;
hipMalloc(&N_ptr, XSIZE*YSIZE);
int LD_N = 1;
double *G_ptr = NULL;
hipMalloc(&G_ptr, XSIZE*YSIZE);
int LD_G = 1;
double *result_ptr = NULL;
hipMalloc(&result_ptr, XSIZE*YSIZE);
int incr = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
read_G_matrix_kernel<<<gridBlock,threadBlock>>>(S,vertex_index,i_index,j_index,is_Bennett,exp_Vj,N_ptr,LD_N,G_ptr,LD_G,result_ptr,incr);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
read_G_matrix_kernel<<<gridBlock,threadBlock>>>(S,vertex_index,i_index,j_index,is_Bennett,exp_Vj,N_ptr,LD_N,G_ptr,LD_G,result_ptr,incr);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
read_G_matrix_kernel<<<gridBlock,threadBlock>>>(S,vertex_index,i_index,j_index,is_Bennett,exp_Vj,N_ptr,LD_N,G_ptr,LD_G,result_ptr,incr);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}