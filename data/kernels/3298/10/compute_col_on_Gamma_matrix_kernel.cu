#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_col_on_Gamma_matrix_kernel(int col_index, int vertex_index, int* indices, double* exp_V, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* col_ptr, int incr) {
// int l = threadIdx.x;
int l = blockIdx.x;

int i_index, j_index;
double delta, exp_Vj;

i_index = indices[l];
j_index = indices[col_index];

exp_Vj = exp_V[col_index];

if (j_index < vertex_index) {
delta = i_index == j_index ? 1 : 0;
col_ptr[l * incr] = (N_ptr[i_index + LD_N * j_index] * exp_Vj - delta) / (exp_Vj - 1.);
}
else
col_ptr[l * incr] = G_ptr[i_index + LD_G * (j_index - vertex_index)];
}