#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void increment_kernel(int *g_data, int inc_value) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
g_data[idx] = g_data[idx] + inc_value;
}