#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simpleMPIKernel(float *input, float *output) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
output[tid] = sqrt(input[tid]);
}