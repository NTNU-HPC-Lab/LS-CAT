#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

int tx = threadIdx.x;
int ty = threadIdx.y;
float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

float2 vterm = *fj;
tx -= r; ty -= r;
float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
vterm.x += s * fx;
vterm.y += s * fy;
*fj = vterm;
}