#include "hip/hip_runtime.h"
#include "includes.h"
/*
* PARA CORRERLO:
*   $ export LD_LIBRARY_PATH=/usr/local/cuda/lib
*   $ export PATH=$PATH:/usr/local/cuda/bin
*   $ nvcc -o matrixTrans matrixTrans.cu -O2 -lc -lm
*   $ ./matrixTrans n
*/

/*
* UNSIGNED INT --> Tipo de dato para enteros, números sin punto decimal.
*                  Los enteros sin signo pueden ser tan grandes como 65535
*                  y tan pequeños como 0.
*                  Son almacenados como 16 bits de información.
*
* SIZE_T --> is an unsigned integer type guaranteed to support the longest
*            object for the platform you use. It is also the result of the
*            sizeof operator.sizeof returns the size of the type in bytes.
*            So in your context of question in both cases you pass a
*            size_t to malloc.
*/

#define NUMBER_THREADS 32

float elapsed_time_ms;
int gpudev = 1;

char *dev_mat_in, *dev_mat_out;

//---------------------------------------------------------------------------

__global__ void kernelTransposeMatrix(const char *mat_in, char *mat_out, unsigned int rows, unsigned int cols){
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

if (idx < cols && idy < rows) {
unsigned int pos = idy * cols + idx;
unsigned int trans_pos = idx * rows + idy;
mat_out[trans_pos] = mat_in[pos];
}
}