#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
x[id] = 1 - exp(-x[id] / lambda);
}
}