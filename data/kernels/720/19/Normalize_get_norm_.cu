#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size023) {
int dim23 = id % size23;
int dim0 = id / size23;

float sum = 0.0;
for (int dim1 = 0; dim1 < size1; dim1++) {
float x = input[(dim0 * size1 + dim1) * size23 + dim23];
sum += x * x;
}
norm[dim0 * size23 + dim23] = sum + 1e-5;
}
}