#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_DIM_X	16
#define BLOCK_DIM_Y	32


#define N	16

__global__ static void KerBitReversalMatrixRow(float *d_lpDstRe, float *d_lpDstIm, float *d_lpSrcRe, float *d_lpSrcIm, int width, int log2x)
{
register int x = blockDim.x * blockIdx.x + threadIdx.x;
register int y = blockDim.y * blockIdx.y + threadIdx.y;
//	int width = 1 << log2x;

if(x < (1 << log2x))
//	for(int i = 0; i < length; i ++)
{
register int index	= 0;
register int t	= x;

for(int j = 0; j < log2x; j ++)
{
index = (index << 1) | (t & 1);
t >>= 1;
}

if(x >= index)
{
register int idx	= width * y + x;
register int jdx	= width * y + index;

register double	 tmpRe	= d_lpDstRe[idx];
register double	 tmpIm	= d_lpDstIm[idx];

d_lpDstRe[idx]	= d_lpSrcRe[jdx];
d_lpDstIm[idx]	= d_lpSrcIm[jdx];

d_lpDstRe[jdx]	= tmpRe;
d_lpDstIm[jdx]	= tmpIm;
}
}
}