#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_DIM_X	16
#define BLOCK_DIM_Y	32


#define N	16

__global__ static void KerBitReversalMatrixCol(float *d_lpDstRe, float *d_lpDstIm, float *d_lpSrcRe, float *d_lpSrcIm, int width, int log2y)
{
register int x = blockDim.x * blockIdx.x + threadIdx.x;
register int y = blockDim.y * blockIdx.y + threadIdx.y;
//	int height = 1 << log2y;

if(y < (1 << log2y))
//	for(int i = 0; i < length; i ++)
{
register int index	= 0;
register int t	= y;

for(int j = 0; j < log2y; j ++)
{
index = (index << 1) | (t & 1);
t >>= 1;
}

if(y >= index)
{
register int idx	= width * y + x;
register int jdx	= width * index + x;

register double	 tmpRe	= d_lpDstRe[idx];
register double	 tmpIm	= d_lpDstIm[idx];

d_lpDstRe[idx]	= d_lpSrcRe[jdx];
d_lpDstIm[idx]	= d_lpSrcIm[jdx];

d_lpDstRe[jdx]	= tmpRe;
d_lpDstIm[jdx]	= tmpIm;
}
}
}