#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullToCOO(int numElem, float* H_vals, double* hamilValues, int dim)
{

int i = threadIdx.x + blockDim.x*blockIdx.x;

if (i < numElem)
{

hamilValues[i] = H_vals[i];


}
}