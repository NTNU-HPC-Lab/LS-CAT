#include "hip/hip_runtime.h"
#include "includes.h"

#define bufSize 700000


struct timeval startwtime,endwtime;

float *h_a;			// Table at host
float *d_a;			// Table at device
int tsize=0;		// number of rows or columns
size_t size = 0 ;	// size of table( tsize* tsize * sizeof(float*))
float* test;

void print(float *);
void make_table();
void serial();
void check();
void copytables();



__global__ void Kernel1(float *A,int N,int k){

int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;
//printf("Hello from %d %d \n",threadIdx.x,threadIdx.y);
if ( A[i*N+j] > A[i*N+k] + A[k*N+j] ){
A[i*N+j] = A[i*N+k] + A[k*N+j];
}
}