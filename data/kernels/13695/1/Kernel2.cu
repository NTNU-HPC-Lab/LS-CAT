#include "hip/hip_runtime.h"
#include "includes.h"

#define bufSize 700000


struct timeval startwtime,endwtime;

float *h_a;			// Table at host
float *d_a;			// Table at device
int tsize=0;		// number of rows or columns
size_t size = 0 ;	// size of table( tsize* tsize * sizeof(float*))
float* test;

void print(float *);
void make_table();
void serial();
void check();
void copytables();



__global__ void Kernel2(float *A,int N,int k){

int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

__shared__ float k_k1,k1_k;
/*	example:
*	if we have to go from D -> F throw k and then k+1 we have to do:
*	DkF check
*  D(k+1)F check
*	Dk(k+1)F check
*	D(k+1)kF check
*	the min of these is the min dist.
*/
if(threadIdx.x==0 && threadIdx.y==0){
k_k1=A[k*N+(k+1)];
k1_k=A[(k+1)*N+k];
}
float x,y,asked,xn,yn;

asked=A[i*N+j];

x=A[k*N+j];
y=A[i*N+k];

// DkF
if(asked>x+y){
asked=x+y;
}

xn=A[i*N+(k+1)];
yn=A[(k+1)*N+j];

__syncthreads();

//	D(k+1)
if(xn>y+k_k1){
xn=y+k_k1;
}
//	(k+1)F
if(yn>x+k1_k){
yn=x+k1_k;
}
//	D(k+1)F or D(k+1)kF or Dk(k+1)F
if(asked>xn+yn){
asked=xn+yn;
}
//	min dist
A[i*N+j]=asked;
}