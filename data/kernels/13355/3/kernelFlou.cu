#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFlou(unsigned char * ptr, unsigned int * debug)
{
__shared__ char sum[4];

sum[0] = 0;
sum[1] = 0;
sum[2] = 0;
sum[3] = 0;

int x = blockIdx.x;
int y = blockIdx.y;
int cc = threadIdx.z;

int index_ptr = (x * DIM_2 + threadIdx.x + (y * DIM_2 + threadIdx.y) * (gridDim.x * DIM_2)) * 4;
int index_avg = (x + y * gridDim.x) * 4;

__syncthreads();

sum[cc] += ptr[index_ptr + cc] / (DIM_2 * DIM_2);

__syncthreads();

ptr[index_ptr + cc] = sum[cc];
debug[index_avg + cc] = sum[cc];
}