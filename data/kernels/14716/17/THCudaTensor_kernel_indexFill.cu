#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexFill( float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val )
{
int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

long flat_size = tensor_size / idx_size;

if (thread_idx < flat_size)
{
long coeff = 0;
for (int i=0; i<idx_size; i++)
{
int leftover = thread_idx;
int srcIdx = 0;
for (int d=0; d<src_nDim; d++)
{
if (d < dim)
{
coeff = leftover / (stride[d] / size_dim);
leftover -= coeff * (stride[d] / size_dim);
srcIdx += coeff * stride[d];
}
else if (d > dim)
{
coeff = leftover / stride[d];
leftover -= coeff * stride[d];
srcIdx += coeff * stride[d];
}
}
tensor[srcIdx + (long)((index[i])-1)*stride[dim]] = val;
}
}
}