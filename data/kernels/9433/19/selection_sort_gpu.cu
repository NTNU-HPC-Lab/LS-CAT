#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
int batch_index = blockIdx.x;
dist+=m*n*batch_index;
idx+=m*k*batch_index;
val+=m*k*batch_index;

int index = threadIdx.x;
int stride = blockDim.x;

float *p_dist;
for (int j=index;j<m;j+=stride) {
p_dist = dist+j*n;
// selection sort for the first k elements
for (int s=0;s<k;++s) {
int min=s;
// find the min
for (int t=s+1;t<n;++t) {
if (p_dist[t]<p_dist[min]) {
min = t;
}
}
// update idx and val
idx[j*n+s] = min;
val[j*n+s] = p_dist[min];
// swap min-th and i-th element
float tmp = p_dist[min];
p_dist[min] = p_dist[s];
p_dist[s] = tmp;
}
}
}