#include "hip/hip_runtime.h"
#include "includes.h"


#define O_Tile_Width 3
#define Mask_width 3
#define width 5
#define Block_width (O_Tile_Width+(Mask_width-1))
#define Mask_radius (Mask_width/2)



__global__ void convolution_1D_tiled(float *N,float *M,float *P)
{
int index_out_x=blockIdx.x*O_Tile_Width+threadIdx.x;
int index_in_x=index_out_x-Mask_radius;
__shared__ float N_shared[Block_width];
float Pvalue=0.0;

//Load Data into shared Memory (into TILE)
if((index_in_x>=0)&&(index_in_x<width))
{
N_shared[threadIdx.x]=N[index_in_x];
}
else
{
N_shared[threadIdx.x]=0.0f;
}
__syncthreads();

//Calculate Convolution (Multiply TILE and Mask Arrays)
if(threadIdx.x<O_Tile_Width)
{
//Pvalue=0.0f;
for(int j=0;j<Mask_width;j++)
{
Pvalue+=M[j]*N_shared[j+threadIdx.x];
}
P[index_out_x]=Pvalue;
}


}