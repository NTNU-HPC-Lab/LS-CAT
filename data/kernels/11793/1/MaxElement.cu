#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void MaxElement(double* Mtr, int Size, int i, int*strnum)
{
double MaxValue=Mtr[i*Size+i];
*strnum=i;

for(int k=i; k<Size; k++)
{

if(fabs(Mtr[i*(Size)+k])>fabs(MaxValue))
{
*strnum=*strnum+1;    //ýòî äëÿ êîìïèëÿòîðà ÷åêåðà
*strnum=k;
MaxValue=Mtr[i*(Size)+k];
}
}

if(fabs(MaxValue)<MINVAL)   //åñëè ìàêñèìàëüíûé ýëåìåíò íèæå ïîðîãîâîãî çíà÷åíèÿ, òî âîçâðàùàåì -1 -> îïðåäåëèòåëü ðàâåí 0 è âûõîäèì èç öèêëà
{
*strnum=-1;
}

}