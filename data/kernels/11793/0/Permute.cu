#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void Permute(double* Dev_Mtr, int* i, int* k, int* Dev_size)
{
int index=blockDim.x*blockIdx.x+threadIdx.x;

if(index<*Dev_size)
{
double tmp=Dev_Mtr[index*(*Dev_size)+(*i)];
Dev_Mtr[index*(*Dev_size)+(*i)]=Dev_Mtr[index*(*Dev_size)+(*k)];
Dev_Mtr[index*(*Dev_size)+(*k)]=tmp;
}

}