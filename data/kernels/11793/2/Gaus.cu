#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void Gaus(double* Mtr, int Size, int i)
{
int index=blockIdx.x*blockDim.x+threadIdx.x;
if(index>i && index< Size)
{
double particial = -Mtr[i*Size+index]/Mtr[i*Size+i];

for(int z=i; z<Size; z++)
{
Mtr[z*Size+index]=Mtr[z*Size+index]+Mtr[z*Size+i]*particial;
}

}
}