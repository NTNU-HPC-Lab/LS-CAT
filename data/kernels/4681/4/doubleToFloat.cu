#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void doubleToFloat(double* input, float* output, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < numElements)
{
output[i] = (float)input[i];
}
}