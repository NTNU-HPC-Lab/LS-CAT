#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floatToDouble(float* input, double* output, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < numElements)
{
output[i] = (double)input[i];
}
}