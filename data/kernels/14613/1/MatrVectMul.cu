#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrVectMul(int *d_c, int *d_a, int *d_b)
{
int i = blockIdx.x*blockDim.x+threadIdx.x;
if(i<N)
{
d_c[i]=0;
for (int k=0;k<N;k++)
d_c[i]+=d_a[i+k*N]*d_b[k];
}
}