#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernal1(int *A, int *B, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < numElements)
B[i] = A[i]+B[i];
}