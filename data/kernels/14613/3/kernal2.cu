#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernal2(int *A, int k, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < numElements)
A[i] = A[i]*k;
}