#include "hip/hip_runtime.h"
#include "includes.h"
#define N 15000

using namespace std;
//Çäåñü: threadIdx.x  èäåíòèôèêàòîð ïîòîêà â áëîêå ïî êîîðäèíàòå x,
//blockIdx.x  èäåíòèôèêàòîð áëîêà â ãðèäå ïî êîîðäèíàòå x,
//blockDim.x  êîëè÷åñòâî ïîòîêîâ â îäíîì áëîêå.

__global__ void MatrVectMul(int *d_c, int *d_a, int *d_b)
{
int i = blockIdx.x*blockDim.x+threadIdx.x;
if(i<N)
{
d_c[i]=0;
for (int k=0;k<N;k++)
d_c[i]+=d_a[i+k*N]*d_b[k];
}
}