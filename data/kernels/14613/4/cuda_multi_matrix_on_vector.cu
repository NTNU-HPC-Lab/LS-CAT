#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_multi_matrix_on_vector(int *matrix, int *vector, int *new_vector, int numElements){
__shared__ int cache[threadsPerBlock];
const int idx = blockDim.x*blockIdx.x + threadIdx.x;//глобальный индекс
const int tIdx = threadIdx.x;//индекс нити
const int k = (numElements - 1 + threadsPerBlock) / threadsPerBlock;//всего кол-во блоков

for (int i = 0; i < k; i++){//в блок влезает threadsPerBlock нитей. Чтобы посчитать всю строку на нужно читать кусок вектора k раз
if (tIdx+threadsPerBlock*i < numElements){//если индекс нити плюс потоковое смещение меньше n то копируем в память shared
cache[tIdx] = vector[tIdx + threadsPerBlock * i];
}
__syncthreads();

int min = numElements - i*threadsPerBlock;//определяем хвост
if (min > threadsPerBlock)min = threadsPerBlock;//если хвост слишком длинный то берём по нитям
if (idx < numElements){
for (int j= 0; j < min; j++){
new_vector[idx] += cache[j]*matrix[(i*threadsPerBlock + j)*numElements + idx];//каждая нить считает свой вектор умножая кусок вектора на сообверствующий кусок матрицы
}
}
__syncthreads();
}
}