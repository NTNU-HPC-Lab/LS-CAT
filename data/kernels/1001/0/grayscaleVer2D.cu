#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscaleVer2D(uchar3* input, uchar3* output, int imageWidth, int imageHeight){
int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
if(tid_x > imageWidth || tid_y > imageHeight) return;
int tid = (int)(tid_x + tid_y * imageWidth);
output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
output[tid].z = output[tid].y = output[tid].x;
}