#include "hip/hip_runtime.h"
#include "includes.h"


#define BLKX 32
#define BLKY 32

hipStream_t gstream;


__global__ void initData(int nbLines, int M, double *h, double *g)
{
long idX = threadIdx.x + blockIdx.x * blockDim.x;

if (idX > nbLines * M)
return;

h[idX] = 0.0L;
g[idX] = 0.0L;
if ( idX >= M +1  && idX  < 2*M-1 ){
h[idX] = 100.0;
g[idX] = 100.0;
}
}