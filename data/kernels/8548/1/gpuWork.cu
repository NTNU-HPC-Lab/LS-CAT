#include "hip/hip_runtime.h"
#include "includes.h"


#define BLKX 32
#define BLKY 32

hipStream_t gstream;


__global__ void gpuWork(double *g, double *h, double *error,  int M, int nbLines){

// This moves thread (0,0) to position (1,1) on the grid
long idX = threadIdx.x + blockIdx.x * blockDim.x +1;
long idY = threadIdx.y + blockIdx.y * blockDim.y +1;
long threadId = threadIdx.y * blockDim.x + threadIdx.x;
long tidX = threadIdx.x + blockIdx.x * blockDim.x;
long tidY = threadIdx.y + blockIdx.y * blockDim.y;

register double temp;
long xSize = M+2;

__shared__ double errors[BLKX*BLKY];

errors[threadId] = 0.0;

if (tidX < M && tidY < nbLines ){
temp = 0.25*(h[(idY-1)*xSize +idX]
+h[((idY+1)*xSize)+idX]
+h[(idY*xSize)+idX-1]
+h[(idY*xSize)+idX+1]);
errors[threadId] = fabs(temp - h[(idY*xSize)+idX]);
g[(idY*xSize)+idX] = temp;
}
else{
return;
}

__syncthreads();


for (unsigned long s = (blockDim.x*blockDim.y)/2; s>0; s=s>>1){
if ( threadId < s ){
errors[threadId] =  fmax(errors[threadId], errors[threadId+s]);
}
__syncthreads();
}


if ( threadId == 0 ){
int id = blockIdx.y * (gridDim.x) + blockIdx.x;
error[id] = errors[0];
}
return;
}