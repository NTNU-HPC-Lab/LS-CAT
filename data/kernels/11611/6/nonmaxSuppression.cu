#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int tex_i(const int * ptData,int y,int x,int step)
{
return ptData[y*step+x];
}
__global__ void nonmaxSuppression(const short2* kpLoc_Device, int count, const int* score_DeviceMat,int cols,int rows,short2* locFinal, float* responseFinal)
{

const int kpIdx = threadIdx.x + blockIdx.x * blockDim.x;

if (kpIdx < count)
{
short2 loc = kpLoc_Device[kpIdx];

int score_Device = tex_i( score_DeviceMat,loc.y, loc.x,cols);

bool ismax =
score_Device > tex_i( score_DeviceMat,loc.y - 1, loc.x - 1,cols) &&
score_Device > tex_i( score_DeviceMat,loc.y - 1, loc.x    ,cols) &&
score_Device > tex_i( score_DeviceMat,loc.y - 1, loc.x + 1,cols) &&

score_Device > tex_i( score_DeviceMat,loc.y    , loc.x - 1,cols) &&
score_Device > tex_i( score_DeviceMat,loc.y    , loc.x + 1,cols) &&

score_Device > tex_i( score_DeviceMat,loc.y + 1, loc.x - 1,cols) &&
score_Device > tex_i( score_DeviceMat,loc.y + 1, loc.x    ,cols) &&
score_Device > tex_i( score_DeviceMat,loc.y + 1, loc.x + 1,cols);

if (ismax)
{
const unsigned int ind = atomicInc(&g_counter, (unsigned int)(-1));

locFinal[ind] = loc;
responseFinal[ind] = static_cast<float>(score_Device);
}
}

}