#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PyrDown_y_g(u_int8_t *ptGrayIn,u_int8_t *ptGrayOut,  int  w, int h)
{


int ix = blockIdx.x*blockDim.x + threadIdx.x;
int iy = blockIdx.y*blockDim.y + threadIdx.y;

if(ix<w && iy<h)// && y>2)
{
float p_2   = ptGrayIn[ix*2+(iy*2-2)*w*2]/16.0f;
float p_1   = ptGrayIn[ix*2+(iy*2-1)*w*2]/4.0f;
float p0    = 3.0f*ptGrayIn[ix*2+iy*2*w*2]/8.0f;
float pp1   = ptGrayIn[ix*2+(iy*2+1)*w*2]/4.0f;
float pp2   = ptGrayIn[ix*2+(iy*2+2)*w*2]/16.0f;

int output  = p_2 + p_1 + p0 + pp1 + pp2;
ptGrayOut[ix+iy*w] = min(output,255);
}
}