#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float *A,float *B,float *C,int numARows,int numAColumns,int numBRows,int numBColumns,int numCRows,int numCColumns)
{
// variable declarations
int row=blockIdx.y * blockDim.y + threadIdx.y;
int col=blockIdx.x * blockDim.x + threadIdx.x;
// code
if((row < numARows) && (col < numBColumns))
{
float Cvalue=0.0;
for(int k=0; k < numAColumns; k++)
{
Cvalue +=A[row * numAColumns + k] * B[k * numBColumns + col];
}
C[row * numCColumns + col]=Cvalue;
}
}