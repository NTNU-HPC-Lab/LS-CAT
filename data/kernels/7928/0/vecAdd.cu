#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float *in1,float *in2,float *out,int len)
{
// variable declarations
int i=blockIdx.x * blockDim.x + threadIdx.x;
// code
if(i < len)
{
out[i]=in1[i]+in2[i];
}
}