#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void trivial_map(int* inp_d, MyInt4* inp_lift, int inp_size) {
const unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
if(gid < inp_size) {
int el = inp_d[gid];
MyInt4 res(el,el,el,el);
if(el < 0) { res.x = 0;  res.y = 0;  res.z = 0; }
inp_lift[gid] = res;
}
}