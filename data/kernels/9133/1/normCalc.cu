#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normCalc (float *d_A, float *d_B, int n) {
int col = blockIdx.x * blockDim.x + threadIdx.x;
__shared__ int row, mu, sigma;
if (col < n){
mu = (float)0.0;
for (row=0; row < n; row++)
mu += d_A[col*n+row];
mu /= (float) n;

__syncthreads();

sigma = (float)0.0;
for (row=0; row < n; row++)
sigma += powf(d_A[col*n+row] - mu, (float)2.0);
sigma /= (float) n;

__syncthreads();

sigma = sqrt((float)sigma);


for (row=0; row < n; row++) {
if (sigma == (float)0.0)
d_B[row*n+col] = (float)0.0;
else
d_B[row*n+col] = (d_A[col*n+row] - mu) / sigma;
}
}
}