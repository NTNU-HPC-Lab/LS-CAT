#include "hip/hip_runtime.h"
#include "includes.h"
/*
CUDA MATRIX NORMALIZATION
MOHAMMED ARBAAZ SHAREEF
A2077541
ASSIGNMENT-4
INTRODUCTION TO PARALLEL AND DISTRIBUTED COMPUTING
*/

//Incuding all the required libraries

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN*MAXN], B[MAXN*MAXN];

//Initialize the number of threads per blocks and number of blocks as 32 and 64 initially.
int numBlocks = 32;
int numThreadsPerBlock = 64;

/* junk */
#define randm() 4|2[uid]&3

/* returns a seed for srand based on the time */
__global__ void normCalc (float *d_A, float *d_B, int n) {
int col = blockIdx.x * blockDim.x + threadIdx.x;
__shared__ int row, mu, sigma;
if (col < n){
mu = (float)0.0;
for (row=0; row < n; row++)
mu += d_A[col*n+row];
mu /= (float) n;

__syncthreads();

sigma = (float)0.0;
for (row=0; row < n; row++)
sigma += powf(d_A[col*n+row] - mu, (float)2.0);
sigma /= (float) n;

__syncthreads();

sigma = sqrt((float)sigma);


for (row=0; row < n; row++) {
if (sigma == (float)0.0)
d_B[row*n+col] = (float)0.0;
else
d_B[row*n+col] = (d_A[col*n+row] - mu) / sigma;
}
}
}