#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ int is_between(float a, float b, float c) {
#if between_method == 0
return (b > a) != (c > a);
#elif between_method == 1
return ((b <= a) && (c > a)) || ((b > a) && (c <= a));
#elif between_method == 2
return ((a - b) == 0.0f) || ((a - b) * (a - c) < 0.0f);
#elif between_method == 3
//Interestingly enough method 3 exactly the same as method 2, only in a different order.
//the performance difference between method 2 and 3 can be huge depending on all the other optimization parameters.
return ((a - b) * (a - c) < 0.0f) || (a - b == 0.0f);
#endif
}
__global__ void cn_pnpoly(int* bitmap, float2* points, int n) {
int i = blockIdx.x * block_size_x * tile_size + threadIdx.x;
if (i < n) {

int c[tile_size];
float2 lpoints[tile_size];
#pragma unroll
for (int ti=0; ti<tile_size; ti++) {
c[ti] = 0;
if (i+block_size_x*ti < n) {
lpoints[ti] = points[i+block_size_x*ti];
}
}

int k = VERTICES-1;

for (int j=0; j<VERTICES; k = j++) {    // edge from vj to vk
float2 vj = d_vertices[j];
float2 vk = d_vertices[k];

#if use_precomputed_slopes == 0
float slope = (vk.x-vj.x) / (vk.y-vj.y);
#elif use_precomputed_slopes == 1
float slope = d_slopes[j];
#endif

#pragma unroll
for (int ti=0; ti<tile_size; ti++) {

float2 p = lpoints[ti];

#if use_method == 0
if (  is_between(p.y, vj.y, vk.y) &&         //if p is between vj and vk vertically
(p.x < slope * (p.y-vj.y) + vj.x)
) {  //if p.x crosses the line vj-vk when moved in positive x-direction
c[ti] = !c[ti];
}

#elif use_method == 1
//Same as method 0, but attempts to reduce divergence by avoiding the use of an if-statement.
//Whether this is more efficient is data dependent because there will be no divergence using method 0, when none
//of the threads within a warp evaluate is_between as true
int b = is_between(p.y, vj.y, vk.y);
c[ti] += b && (p.x < vj.x + slope * (p.y - vj.y));

#endif


}

}

#pragma unroll
for (int ti=0; ti<tile_size; ti++) {
//could do an if statement here if 1s are expected to be rare
if (i+block_size_x*ti < n) {
#if use_method == 0
bitmap[i+block_size_x*ti] = c[ti];
#elif use_method == 1
bitmap[i+block_size_x*ti] = c[ti] & 1;
#endif
}
}
}

}