#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_kernel(float *C, float *A, float *B) {

__shared__ float sA[block_size_y*tile_size_y][block_size_x];
__shared__ float sB[block_size_y*tile_size_y][block_size_x * tile_size_x];

int tx = threadIdx.x;
int ty = threadIdx.y;
int x = blockIdx.x * block_size_x * tile_size_x + threadIdx.x;
int y = blockIdx.y * block_size_y * tile_size_y + threadIdx.y;
int k, kb;

float sum[tile_size_y][tile_size_x];
#pragma unroll
for (int i = 0; i < tile_size_y; i++) {
#pragma unroll
for (int j = 0; j < tile_size_x; j++) {
sum[i][j] = 0.0f;
}
}

for (k = 0; k < WIDTH; k += block_size_x) {

__syncthreads();
#pragma unroll
for (int i = 0; i < tile_size_y; i++) {
sA[ty + block_size_y * i][tx] = A[(y+i*block_size_y) * WIDTH + k + tx];

#pragma unroll
for (int j = 0; j < tile_size_x; j++) {
sB[ty + block_size_y * i][tx + j * block_size_x] = B[(k + ty + block_size_y * i) * WIDTH + x + j * block_size_x];
}
}
__syncthreads();

//compute
#pragma unroll
for (kb = 0; kb < block_size_x; kb++) {

#pragma unroll
for (int i = 0; i < tile_size_y; i++) {
#pragma unroll
for (int j = 0; j < tile_size_x; j++) {
sum[i][j] += sA[ty + block_size_y * i][kb] * sB[kb][tx + j * block_size_x];
}
}

}

}

//store result
#pragma unroll
for (int i = 0; i < tile_size_y; i++) {
#pragma unroll
for (int j = 0; j < tile_size_x; j++) {
C[y * WIDTH + x + block_size_y * i * WIDTH + j * block_size_x] = sum[i][j];
}
}

}