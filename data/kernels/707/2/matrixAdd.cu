#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd(float *A, float *B, float *C, int n)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int num = n;

int i = row * num + col;

if (row < num && col < num)
{
C[i] = A[i] + B[i];
}
}