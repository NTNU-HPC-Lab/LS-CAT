#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imgBlur(float* imgIn, float* imgOut, int imageWidth, int imageHeight)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;

if(idx<imageWidth && idy<imageHeight)
{
float sum = imgIn[idx*imageWidth+idy];

if(idx>0 && idy>0)
sum += imgIn[(idx-1)*imageWidth+(idy-1)];

if(idx>0)
sum += imgIn[(idx-1)*imageWidth+idy];

if(idx<imageWidth-1)
sum += imgIn[(idx+1)*imageWidth+idy];

if(idx<imageWidth-1 && idy<imageHeight-1)
sum += imgIn[(idx+1)*imageWidth+idy+1];

if(idx<imageWidth && idy>0)
sum += imgIn[(idx+1)*imageWidth+idy-1];

if(idy>0)
sum += imgIn[idx*imageWidth+idy-1];

if(idy<imageHeight)
sum += imgIn[idx*imageWidth+idy+1];

if(idx>0 && idy<imageHeight)
sum += imgIn[(idx-1)*imageWidth+idy+1];

imgOut[idx*imageWidth+idy] = sum / (float)(BLUR_SIZE*BLUR_SIZE);

}
}