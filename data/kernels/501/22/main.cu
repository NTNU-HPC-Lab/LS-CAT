#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ftest.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int diagFlag = 1;
int p = 1;
int rows = XSIZE;
int colsx = 1;
int colsy = 1;
int rCols = 1;
int unrCols = 1;
float *obs = NULL;
hipMalloc(&obs, XSIZE*YSIZE);
int obsDim = 1;
float *rCoeffs = NULL;
hipMalloc(&rCoeffs, XSIZE*YSIZE);
int rCoeffsDim = 1;
float *unrCoeffs = NULL;
hipMalloc(&unrCoeffs, XSIZE*YSIZE);
int unrCoeffsDim = 1;
float *rdata = NULL;
hipMalloc(&rdata, XSIZE*YSIZE);
int rdataDim = 1;
float *unrdata = NULL;
hipMalloc(&unrdata, XSIZE*YSIZE);
int unrdataDim = 1;
float *dfStats = NULL;
hipMalloc(&dfStats, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ftest<<<gridBlock,threadBlock>>>(diagFlag,p,rows,colsx,colsy,rCols,unrCols,obs,obsDim,rCoeffs,rCoeffsDim,unrCoeffs,unrCoeffsDim,rdata,rdataDim,unrdata,unrdataDim,dfStats);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ftest<<<gridBlock,threadBlock>>>(diagFlag,p,rows,colsx,colsy,rCols,unrCols,obs,obsDim,rCoeffs,rCoeffsDim,unrCoeffs,unrCoeffsDim,rdata,rdataDim,unrdata,unrdataDim,dfStats);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ftest<<<gridBlock,threadBlock>>>(diagFlag,p,rows,colsx,colsy,rCols,unrCols,obs,obsDim,rCoeffs,rCoeffsDim,unrCoeffs,unrCoeffsDim,rdata,rdataDim,unrdata,unrdataDim,dfStats);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}