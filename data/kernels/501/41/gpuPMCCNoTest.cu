#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__global__ void gpuPMCCNoTest(const float * vectsa, size_t na, const float * vectsb, size_t nb, size_t dim, const float * numPairs, const float * means, const float * sds, float * correlations)
{
size_t
offset, stride,
x = blockIdx.x, y = blockIdx.y,
tx = threadIdx.x;
float
a, b, n, scoreA, scoreB;
__shared__ float
meanA, meanB,
sdA, sdB,
threadSums[NUMTHREADS];

if((x >= na) || (y >= nb))
return;

if(tx == 0) {
meanA = means[x*nb*2+y*2];
meanB = means[x*nb*2+y*2+1];
sdA = sds[x*nb*2+y*2];
sdB = sds[x*nb*2+y*2+1];
n = numPairs[x*nb+y];
}
__syncthreads();

threadSums[tx] = 0.f;
for(offset = tx; offset < dim; offset += NUMTHREADS) {
a = vectsa[x * dim + offset];
b = vectsb[y * dim + offset];

scoreA = (a - meanA) / sdA;
scoreB = (b - meanB) / sdB;
threadSums[tx] += scoreA * scoreB;
}
__syncthreads();

for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
if(tx < stride) threadSums[tx] += threadSums[tx + stride];
__syncthreads();
}
if(tx == 0) correlations[x*nb+y] = threadSums[0] / (n - 1.f);
}