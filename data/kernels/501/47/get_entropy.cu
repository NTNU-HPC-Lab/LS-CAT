#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_entropy(int nbins, int nsamples, int nx, float * bin_scores, int pitch_bin_scores, float * entropies)
{
int
col_x = blockDim.x * blockIdx.x + threadIdx.x;

if(col_x >= nx)
return;

float
* in_col = bin_scores + col_x * pitch_bin_scores,
entropy = 0.f, prob, logp;

for(int i = 0; i < nbins; i++) {
prob = 0.f;
for(int j = 0; j < nsamples; j++)
prob += in_col[j * nbins + i];
prob /= (double) nsamples;

if(prob <= 0.f)
logp = 0.f;
else
logp = __log2f(prob);

entropy += prob * logp;
}
entropies[col_x] = -entropy;
}