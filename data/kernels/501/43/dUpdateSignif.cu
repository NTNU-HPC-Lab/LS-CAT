#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32








__device__ int dIsSignificant(float signif, int df) {
float tcutoffs[49] = {
// cuttoffs for degrees of freedom <= 30
637.000, 31.600, 2.920, 8.610, 6.869, 5.959, 5.408, 5.041, 4.781,
4.587, 4.437, 4.318, 4.221, 4.140, 4.073, 4.015, 3.965, 3.922,
3.883, 3.850, 3.819, 3.792, 3.768, 3.745, 3.725, 3.707, 3.690,
3.674, 3.659, 3.646,
// cuttoffs for even degrees of freedom > 30 but <= 50
3.622, 3.601, 3.582, 3.566, 3.551, 3.538, 3.526, 3.515, 3.505, 3.496,
// 55 <= df <= 70 by 5s
3.476, 3.460, 3.447, 3.435,
3.416, // 80
3.390, // 100
3.357, // 150
3.340, // 200
3.290  // > 200
};

size_t index = 0;
if(df <= 0) return 0;
else if(df <= 30) index = df - 1;
else if(df <= 50) index = 30 + (df + (df%2) - 32) / 2;
else if(df <= 70) {
if(df <= 55) index = 40;
else if(df <= 60) index = 41;
else if(df <= 65) index = 42;
else if(df <= 70) index = 43;
}
else if(df <= 80) index = 44;
else if(df <= 100) index = 45;
else if(df <= 150) index = 46;
else if(df <= 200) index = 47;
else if(df > 200) index = 48;

if(fabsf(signif) < tcutoffs[index]) return FALSE;

return TRUE;
}
__global__ void dUpdateSignif(const float * gpuData, size_t n, float * gpuResults)
{
size_t
i, start, inrow, outrow,
bx = blockIdx.x, tx = threadIdx.x;
float
radicand, cor, npairs, tscore;

start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;

for(i = 0; i < THREADWORK; i++) {
if(start+i > n) break;

inrow = (start+i)*5;
outrow = (start+i)*6;

cor = gpuData[inrow+3];
npairs = gpuData[inrow+4];

if(cor >= 0.999)
tscore = 10000.0;
else {
radicand = (npairs - 2.f) / (1.f - cor * cor);
tscore = cor * sqrtf(radicand);
}
if(dIsSignificant(tscore, (int)npairs)) {
gpuResults[outrow]   = gpuData[inrow];
gpuResults[outrow+1] = gpuData[inrow+1];
gpuResults[outrow+2] = gpuData[inrow+2];
gpuResults[outrow+3] = cor;
gpuResults[outrow+4] = tscore;
gpuResults[outrow+5] = npairs;
} else {
gpuResults[outrow] = -1.f;
}
}
}