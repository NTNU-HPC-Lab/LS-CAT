#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512





// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void gpuFindMax(int n, float * data, int threadWorkLoad, int * maxIndex)
{
int
j, k,
start = threadWorkLoad * threadIdx.x,
end = start + threadWorkLoad;
__shared__ int maxIndicies[NTHREADS];

maxIndicies[threadIdx.x] = -1;

if(start >= n)
return;

int localMaxIndex = start;
for(int i = start+1; i < end; i++) {
if(i >= n)
break;
if(data[i] > data[localMaxIndex])
localMaxIndex = i;
}
maxIndicies[threadIdx.x] = localMaxIndex;
__syncthreads();

for(int i = blockDim.x >> 1; i > 0; i >>= 1) {
if(threadIdx.x < i) {
j = maxIndicies[threadIdx.x];
k = maxIndicies[i + threadIdx.x];
if((j != -1) && (k != -1) && (data[j] < data[k]))
maxIndicies[threadIdx.x] = k;
}
__syncthreads();
}
if(threadIdx.x == 0) {
*maxIndex = maxIndicies[0];
// debug printing
// printf("max index: %d\n", *maxIndex);
// printf("max norm: %f\n", data[*maxIndex]);
// end debug printing
}
}