#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32














__global__ void dot_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

// If all element is to be computed
if(x < n_a && y < n_b) {
__shared__ float temp[NUM_THREADS];

temp[threadIdx.x] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
float t = vg_a[x * pitch_a + offset] * vg_b[y * pitch_b + offset];
temp[threadIdx.x] += t;
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
d[y * pitch_d + x] = temp[0];
}
}
}