#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float do_fraction(float numer, float denom) {
float result = 0.f;

if((numer == denom) && (numer != 0.f))
result = 1.f;
else if(denom != 0.f)
result = numer / denom;

return result;
}
__global__ void get_bin_scores(int nbins, int order, int nknots, float * knots, int nsamples, int nx, float * x, int pitch_x, float * bins, int pitch_bins)
{
int
col_x = blockDim.x * blockIdx.x + threadIdx.x;

if(col_x >= nx)
return;

float
ld, rd, z,
term1, term2,
* in_col = x + col_x * pitch_x,
* bin_col = bins + col_x * pitch_bins;
int i0;

for(int k = 0; k < nsamples; k++, bin_col += nbins) {
z = in_col[k];
i0 = (int)floorf(z) + order - 1;
if(i0 >= nbins)
i0 = nbins - 1;

bin_col[i0] = 1.f;
for(int i = 2; i <= order; i++) {
for(int j = i0 - i + 1; j <= i0; j++) {
rd = do_fraction(knots[j + i] - z, knots[j + i] - knots[j + 1]);

if((j < 0) || (j >= nbins) || (j >= nknots) || (j + i - 1 < 0) || (j > nknots))
term1 = 0.f;
else {
ld = do_fraction(z - knots[j],
knots[j + i - 1] - knots[j]);
term1 = ld * bin_col[j];
}

if((j + 1 < 0) || (j + 1 >= nbins) || (j + 1 >= nknots) || (j + i < 0) || (j + i >= nknots))
term2 = 0.f;
else {
rd = do_fraction(knots[j + i] - z,
knots[j + i] - knots[j + 1]);
term2 = rd * bin_col[j + 1];
}
bin_col[j] = term1 + term2;
}
}
}
}