#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_add(double const *A, double const *B, double *C, int const N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
// if(i%512==0)
//     printf("index %d\n",i);
if (i < N)
C[i] = A[i] + B[i];
}