#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultKernel (float *d_A, float *d_B, float *d_C, int N)
{
// Calculate the row index of the d_C element and d_A
int row = blockIdx.y * blockDim.y + threadIdx.y;

// Calculate the column index of d_C and d_B
int col = blockIdx.x * blockDim.x + threadIdx.x;

if ((row < N) && (col < N))
{
float Cvalue = 0;
for (int k = 0; k < N; k++)
Cvalue += d_A[row * N + k] * d_B[k * N + col];
d_C[row * N + col] = Cvalue;
}
}