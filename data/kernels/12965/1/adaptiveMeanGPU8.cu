#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptiveMeanGPU8 (float* D, int32_t D_width, int32_t D_height) {

// Global coordinates and Pixel id
uint32_t u0 = blockDim.x*blockIdx.x + threadIdx.x + 4;
uint32_t v0 = blockDim.y*blockIdx.y + threadIdx.y + 4;
uint32_t idx = v0*D_width + u0;
//Local thread coordinates
uint32_t ut = threadIdx.x + 4;
uint32_t vt = threadIdx.y + 4;

//If out of filter range return instantly
if(u0 > (D_width - 4) || v0 > (D_height - 4))
return;

//Allocate Shared memory array with an appropiate margin for the bitlateral filter
//Since we are using 8 pixels with the center pixel being 5,
//we need 4 extra on left and top and 3 extra on right and bottom
__shared__ float D_shared[32+7][32+7];
//Populate shared memory
if(threadIdx.x == blockDim.x-1){
D_shared[ut+1][vt] = D[idx+1];
D_shared[ut+2][vt] = D[idx+2];
D_shared[ut+3][vt] = D[idx+3];
//D_shared[ut+4][vt] = D[idx+4];
}
if(threadIdx.x == 0){
D_shared[ut-4][vt] = D[idx-4];
D_shared[ut-3][vt] = D[idx-3];
D_shared[ut-2][vt] = D[idx-2];
D_shared[ut-1][vt] = D[idx-1];
}
if(threadIdx.y == 0){
D_shared[ut][vt-4] = D[(v0-4)*D_width+u0];
D_shared[ut][vt-3] = D[(v0-3)*D_width+u0];
D_shared[ut][vt-2] = D[(v0-2)*D_width+u0];
D_shared[ut][vt-1] = D[(v0-1)*D_width+u0];
}
if(threadIdx.y == blockDim.y-1){
D_shared[ut][vt+1] = D[(v0+1)*D_width+u0];
D_shared[ut][vt+2] = D[(v0+2)*D_width+u0];
D_shared[ut][vt+3] = D[(v0+3)*D_width+u0];
//D_shared[ut][vt+4] = D[(v0+4)*D_width+u0];
}

if(D[idx] < 0){
// zero input disparity maps to -10 (this makes the bilateral
// weights of all valid disparities to 0 in this region)
D_shared[ut][vt] = -10;
}else{
D_shared[ut][vt] = D[idx];
}
__syncthreads();

// full resolution: 8 pixel bilateral filter width
// D(x) = sum(I(xi)*f(I(xi)-I(x))*g(xi-x))/W(x)
// W(x) = sum(f(I(xi)-I(x))*g(xi-x))
// g(xi-x) = 1
// f(I(xi)-I(x)) = 4-|I(xi)-I(x)| if greater than 0, 0 otherwise
// horizontal filter

// Current pixel being filtered is middle of our set (4 back, in orginal its 3 for some reason)
//Note this isn't truely the center since original uses 8 vectore resisters
float val_curr = D_shared[ut][vt];

float weight_sum0 = 0;
float weight_sum = 0;
float factor_sum = 0;

for(int32_t i=0; i < 8; i++){
weight_sum0 = 4.0f - fabs(D_shared[ut+(i-4)][vt]-val_curr);
weight_sum0 = max(0.0f, weight_sum0);
weight_sum += weight_sum0;
factor_sum += D_shared[ut+(i-4)][vt]*weight_sum0;
}

if (weight_sum>0) {
float d = factor_sum/weight_sum;
if (d>=0) *(D+idx) = d;
}

__syncthreads();
//Update shared memory
if(threadIdx.x == blockDim.x-1){
D_shared[ut+1][vt] = D[idx+1];
D_shared[ut+2][vt] = D[idx+2];
D_shared[ut+3][vt] = D[idx+3];
//D_shared[ut+4][vt] = D[idx+4];
}
if(threadIdx.x == 0){
D_shared[ut-4][vt] = D[idx-4];
D_shared[ut-3][vt] = D[idx-3];
D_shared[ut-2][vt] = D[idx-2];
D_shared[ut-1][vt] = D[idx-1];
}
if(threadIdx.y == 0){
D_shared[ut][vt-4] = D[(v0-4)*D_width+u0];
D_shared[ut][vt-3] = D[(v0-3)*D_width+u0];
D_shared[ut][vt-2] = D[(v0-2)*D_width+u0];
D_shared[ut][vt-1] = D[(v0-1)*D_width+u0];
}
if(threadIdx.y == blockDim.y-1){
D_shared[ut][vt+1] = D[(v0+1)*D_width+u0];
D_shared[ut][vt+2] = D[(v0+2)*D_width+u0];
D_shared[ut][vt+3] = D[(v0+3)*D_width+u0];
//D_shared[ut][vt+4] = D[(v0+4)*D_width+u0];
}

if(D[idx] < 0){
D_shared[ut][vt] = -10;
}else{
D_shared[ut][vt] = D[idx];
}

__syncthreads();

// vertical filter
// set pixel of interest
val_curr = D_shared[ut][vt];

weight_sum0 = 0;
weight_sum = 0;
factor_sum = 0;

for(int32_t i=0; i < 8; i++){
weight_sum0 = 4.0f - fabs(D_shared[ut][vt+(i-4)]-val_curr);
weight_sum0 = max(0.0f, weight_sum0);
weight_sum += weight_sum0;
factor_sum += D_shared[ut][vt+(i-4)]*weight_sum0;
}

if (weight_sum>0) {
float d = factor_sum/weight_sum;
if (d>=0) *(D+idx) = d;
}

}