#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "findMatch_GPU.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int32_t *u_vals = NULL;
hipMalloc(&u_vals, XSIZE*YSIZE);
int32_t *v_vals = NULL;
hipMalloc(&v_vals, XSIZE*YSIZE);
int32_t size_total = XSIZE*YSIZE;
float *planes_a = NULL;
hipMalloc(&planes_a, XSIZE*YSIZE);
float *planes_b = NULL;
hipMalloc(&planes_b, XSIZE*YSIZE);
float *planes_c = NULL;
hipMalloc(&planes_c, XSIZE*YSIZE);
int32_t *disparity_grid = NULL;
hipMalloc(&disparity_grid, XSIZE*YSIZE);
int32_t *grid_dims = NULL;
hipMalloc(&grid_dims, XSIZE*YSIZE);
uint8_t *I1_desc = NULL;
hipMalloc(&I1_desc, XSIZE*YSIZE);
uint8_t *I2_desc = NULL;
hipMalloc(&I2_desc, XSIZE*YSIZE);
int32_t *P = NULL;
hipMalloc(&P, XSIZE*YSIZE);
int32_t plane_radius = 1;
int32_t width = XSIZE;
int32_t height = YSIZE;
bool *valids = NULL;
hipMalloc(&valids, XSIZE*YSIZE);
bool right_image = 1;
float *D = NULL;
hipMalloc(&D, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
findMatch_GPU<<<gridBlock,threadBlock>>>(u_vals,v_vals,size_total,planes_a,planes_b,planes_c,disparity_grid,grid_dims,I1_desc,I2_desc,P,plane_radius,width,height,valids,right_image,D);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
findMatch_GPU<<<gridBlock,threadBlock>>>(u_vals,v_vals,size_total,planes_a,planes_b,planes_c,disparity_grid,grid_dims,I1_desc,I2_desc,P,plane_radius,width,height,valids,right_image,D);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
findMatch_GPU<<<gridBlock,threadBlock>>>(u_vals,v_vals,size_total,planes_a,planes_b,planes_c,disparity_grid,grid_dims,I1_desc,I2_desc,P,plane_radius,width,height,valids,right_image,D);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}