#include "hip/hip_runtime.h"
#include "includes.h"
__device__ uint32_t getAddressOffsetGrid_GPU (const int32_t& x,const int32_t& y,const int32_t& d,const int32_t& width,const int32_t& disp_num) {
return (y*width+x)*disp_num+d;
}
__device__ uint32_t getAddressOffsetImage_GPU (const int32_t& u,const int32_t& v,const int32_t& width) {
return v*width+u;
}
__global__ void findMatch_GPU (int32_t* u_vals, int32_t* v_vals, int32_t size_total, float* planes_a, float* planes_b, float* planes_c, int32_t* disparity_grid, int32_t *grid_dims, uint8_t* I1_desc, uint8_t* I2_desc, int32_t* P, int32_t plane_radius, int32_t width ,int32_t height, bool* valids, bool right_image, float* D) {

// get image width and height
const int32_t disp_num    = grid_dims[0]-1;
const int32_t window_size = 2;

//TODO: Remove hard code and use param
bool subsampling = false;
bool match_texture = true;
int32_t grid_size = 20;

// Pixel id
uint32_t idx = blockDim.x*blockIdx.x + threadIdx.x;

// Check that we are in range
if(idx >= size_total)
return;

// Else get our values from memory
uint32_t u = u_vals[idx];
uint32_t v = v_vals[idx];
float plane_a = planes_a[idx];
float plane_b = planes_b[idx];
float plane_c = planes_c[idx];
bool valid = valids[idx];

// address of disparity we want to compute
uint32_t d_addr;
if (subsampling) d_addr = getAddressOffsetImage_GPU(u/2,v/2,width/2);
else             d_addr = getAddressOffsetImage_GPU(u,v,width);

// check if u is ok
if (u<window_size || u>=width-window_size)
return;

// compute line start address
int32_t  line_offset = 16*width*max(min(v,height-3),2);
uint8_t *I1_line_addr,*I2_line_addr;
if (!right_image) {
I1_line_addr = I1_desc+line_offset;
I2_line_addr = I2_desc+line_offset;
} else {
I1_line_addr = I2_desc+line_offset;
I2_line_addr = I1_desc+line_offset;
}

// compute I1 block start address
uint8_t* I1_block_addr = I1_line_addr+16*u;

// does this patch have enough texture?
int32_t sum = 0;
for (int32_t i=0; i<16; i++)
sum += abs((int32_t)(*(I1_block_addr+i))-128);
if (sum<match_texture)
return;

// compute disparity, min disparity and max disparity of plane prior
int32_t d_plane     = (int32_t)(plane_a*(float)u+plane_b*(float)v+plane_c);
int32_t d_plane_min = max(d_plane-plane_radius,0);
int32_t d_plane_max = min(d_plane+plane_radius,disp_num-1);

// get grid pointer
int32_t  grid_x    = (int32_t)floor((float)u/(float)grid_size);
int32_t  grid_y    = (int32_t)floor((float)v/(float)grid_size);
uint32_t grid_addr = getAddressOffsetGrid_GPU(grid_x,grid_y,0,grid_dims[1],grid_dims[0]);
int32_t  num_grid  = *(disparity_grid+grid_addr);
int32_t* d_grid    = disparity_grid+grid_addr+1;

// loop variables
int32_t d_curr, u_warp, val;
int32_t min_val = 10000;
int32_t min_d   = -1;

// left image
for (int32_t i=0; i<num_grid; i++) {
d_curr = d_grid[i];
if (d_curr<d_plane_min || d_curr>d_plane_max) { //If the current disparity is out of the planes range
u_warp = u-d_curr+2*right_image*d_curr; //uwarp diffe
if (u_warp<window_size || u_warp>=width-window_size)
continue;
u_warp = 16*u_warp;
val = 0;
for(int j=0; j<16; j++){
//val += abs((int32_t)(*(I1_block_addr+j))-(int32_t)(*(I2_line_addr+j+16*u_warp)));
val = __sad((int)(*(I1_block_addr+j)),(int)(*(I2_line_addr+j+u_warp)),val);
}

if (val<min_val) {
min_val = val;
min_d   = d_curr;
}
}
}
//disparity inside the grid
for (d_curr=d_plane_min; d_curr<=d_plane_max; d_curr++) {
u_warp = u-d_curr+2*right_image*d_curr;
if (u_warp<window_size || u_warp>=width-window_size)
continue;
u_warp = 16*u_warp;
val = 0;
for(int j=0; j<16; j++){
//val += abs((int32_t)(*(I1_block_addr+j))-(int32_t)(*(I2_line_addr+j+16*u_warp)));
val = __sad((int)(*(I1_block_addr+j)),(int)(*(I2_line_addr+j+u_warp)),val);
}
val += valid?*(P+abs(d_curr-d_plane)):0;
if (val<min_val) {
min_val = val;
min_d   = d_curr;
}
}

// set disparity value
if (min_d>=0) *(D+d_addr) = min_d; // MAP value (min neg-Log probability)
else          *(D+d_addr) = -1;    // invalid disparity
}