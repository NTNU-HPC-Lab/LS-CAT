#include "hip/hip_runtime.h"
#include "includes.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Jaron Pollman
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports

// problem size (vector length) N
static const int N = 12345678; //#of threads?

// Number of terms to use when approximating sine
static const int TERMS = 6; //# of blocks

// kernel function (CPU - Do not modify)
__global__ void paralellSine(float *input, float *output)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x; //Proper indexing of elements.
float value = input[idx];
float numer = input[idx] * input[idx] * input[idx];
int denom = 6;
int sign = -1;

for (int j=1; j<=TERMS; j++)
{
value += sign * numer/denom;
numer *= input[idx] * input[idx];
denom *= (2 * j + 2) * (2 * j + 3);
sign *= -1;
}
output[idx] = value;


}