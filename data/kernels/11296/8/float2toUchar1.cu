#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float2toUchar1(float2 *inputImage, uchar1 *outputImage, int width, int height, int index) {
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;
float2 pixelf = inputImage[offset];
float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;
uchar1 pixel;
pixel.x = (unsigned char) pixelfIndexed;
outputImage[offset] = pixel;
}