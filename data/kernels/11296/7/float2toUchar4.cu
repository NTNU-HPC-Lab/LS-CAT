#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float2toUchar4(float2 *inputImage, uchar4 *outputImage, int width, int height, int index) {
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;
float2 pixelf = inputImage[offset];
float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;
uchar4 pixel;
pixel.x = (unsigned char) abs(pixelfIndexed);
pixel.y = (unsigned char) abs(pixelfIndexed);
pixel.z = (unsigned char) abs(pixelfIndexed);
pixel.w = (unsigned char) abs(pixelfIndexed);
outputImage[offset] = pixel;
}