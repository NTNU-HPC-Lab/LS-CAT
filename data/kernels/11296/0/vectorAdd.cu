#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/

typedef struct {
float *hA, *hB, *hC;
float *dA, *dB, *dC;
int element_count;
size_t vector_bytes;
int v_threadsPerBlock;
int v_blocksPerGrid;
hipStream_t stream;
} ThreadContext;

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
C[i] = A[i] + B[i];
}
}