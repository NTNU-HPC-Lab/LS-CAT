#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void GetMatrixA(int* OCTData, float* MatrixA, int NumPolynomial, int OneDataSize)
{
// 這個 Function 是去取得 MatrixA 的值
int id = blockIdx.x * blockDim.x + threadIdx.x;

// 例外判斷 (理論上應該也是不會超過)
if (id >= (NumPolynomial + 1) * (NumPolynomial + 1))
{
printf("多項式 Fitting 有問題!\n");
return;
}

// 算 Index
int rowIndex = id % (NumPolynomial + 1);
int colsIndex = id / (NumPolynomial + 1);

// 做相加
float value = 0;
for (int i = 0; i < OneDataSize; i++)
{
// 抓出兩項的值
float FirstValue = (float)i / OneDataSize;
float SecondValue = (float)i / OneDataSize;
value += pow(FirstValue, NumPolynomial - rowIndex) * pow(SecondValue, NumPolynomial - colsIndex);
}
MatrixA[id] = value;
}