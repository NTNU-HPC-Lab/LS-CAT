#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ShiftFinalData(float* AfterFFTData, float* ShiftData, int SizeX, int SizeY, int FinalSizeZ, int FinalDataSize)
{
// 這邊要做位移
// 由於硬體是這樣子 ↓
// => | ->
// ("->" 是指第一段，"=>" 是指第二段)
int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

if (id >= FinalDataSize)
{
printf("Shift Data 有錯誤!!\n");
return;
}

// 這邊的算法要對應回去原本的資料
int idZ = id % FinalSizeZ;
int tempIndex = id / FinalSizeZ;
int idX = tempIndex % SizeX;
int idY = tempIndex / SizeX;

// SizeY 折回來
// (0 ~ 124 125 ~ 249)
//		↓
// (125 ~ 249 0 ~ 124)
idY = (idY + SizeY / 2) % SizeY;

int NewIndex = idY * SizeX * FinalSizeZ + idX * FinalSizeZ + idZ;
ShiftData[id] = AfterFFTData[NewIndex];
//ShiftData[id] = AfterFFTData[id];
}