#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ZCalcBrightness(float* DataArray, float* BrightArray, int size, int rows, int cols, int startIndex)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= size * rows)						// 超出範圍
return;

// 算 Index
int sizeIndex = id / rows;
int rowIndex = id % rows;

BrightArray[id] = 0;
for (int i = startIndex; i < cols; i++)
{
int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
BrightArray[id] += DataArray[currentID];
}
}