#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void FrequencyAdjust(int* OCTData, float* KSpaceData, float* PXScale, int* IndexArray, int CutIndex, int SizeX, int SizeY, int SizeZ)
{
// 這邊是 Denoise，把兩個 Channel 的資料相加
int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

if (id >= SizeX * SizeY * SizeZ)
{
printf("Frequency 轉換的地方有問題");
return;
}

// 算回原本的 Index
int idZ = id % SizeZ;
if (IndexArray[idZ] == -1 || idZ >= CutIndex || idZ == 0)
{
KSpaceData[id] = 0;
return;
}

// 要算斜率前，先拿出上一筆資料
int LastPXScaleIndex = (IndexArray[idZ] - 1 <= 0 ? 0 : IndexArray[idZ] - 1);

double m = (double)(OCTData[id] - OCTData[id - 1]) / (PXScale[IndexArray[idZ]] - PXScale[LastPXScaleIndex]);
double c = OCTData[id] - m * PXScale[IndexArray[idZ]];
KSpaceData[id] = m * idZ + c;
}