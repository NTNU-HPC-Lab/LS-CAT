#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void MinusByFittingFunction(int* OCTData, float* PolyValue, int SizeZ)
{
// 這邊要減掉 Fitting Data
int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

// 先拿出他是第幾個 Z
int idZ = id % SizeZ;

// 減掉預測的值
OCTData[id] -= PolyValue[idZ];
}