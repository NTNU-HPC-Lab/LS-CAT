#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void CombineTwoChannels_Single(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
// 這邊是 Denoise，把兩個 Channel 的資料相加
int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

// 這邊應該是不會發生，就當作例外判斷
if (id >= SizeX * SizeY * SizeZ)
{
printf("Combine Two Channel 有 Error!\n");
return;
}

int BoxSize = SizeX * SizeZ;										// 這邊沒有反掃，所以直接接上大小
int BoxIndex = id / BoxSize;
int BoxLeft = id % BoxSize;

OCTData[id] = (OCTData_2Channls[BoxIndex * 2 * BoxSize + BoxLeft] +
OCTData_2Channls[(BoxIndex * 2 + 1) * BoxSize + BoxLeft]) / 2;
}