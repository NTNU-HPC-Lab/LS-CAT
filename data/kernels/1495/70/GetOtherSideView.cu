#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void GetOtherSideView(float* Data, float* OtherSideData, int SizeX, int SizeY, int FinalSizeZ)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= SizeX * SizeY)
{
printf("範圍有錯!!\n");
return;
}

// id 換算
int idX = id / SizeY;
int idY = id % SizeY;
int DataOffsetIndex = idX * SizeY * FinalSizeZ + idY * FinalSizeZ;

// 總和一個 SizeZ
float totalZ = 0;
for (int i = 0; i < FinalSizeZ; i++)
totalZ += Data[DataOffsetIndex + i];


// 這邊的單位要調整一下
// rows => 是張樹 (SizeY)
// cols => 是 SizeX
int offsetIndex = idY * SizeX + idX;
OtherSideData[offsetIndex] = totalZ;
}