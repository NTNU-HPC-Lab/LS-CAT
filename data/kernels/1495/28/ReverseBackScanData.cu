#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ReverseBackScanData(int* OCTData, int SizeX, int SizeY, int SizeZ)
{
// 這邊是要反轉 反掃的資料
int id = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +			// Y	=> (Y * 2 + 1) * (2 * 1024)						=> 1, 3, 5, 7, 9
blockIdx.x * gridDim.z * blockDim.x +											// X	=> X * (125 * 2) * (2 * 1024)
blockIdx.z * blockDim.x +														// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

int changeID = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +		// Y	=> (Y * 2 + 1) * (2 * 1024)						=> 1, 3, 5, 7, 9
(gridDim.y * 2 - blockIdx.x - 1) * gridDim.z * blockDim.x +						// X	=> (250 - X - 1) * (125 * 2) * (2 * 1024)
blockIdx.z * blockDim.x +														// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

int value = OCTData[id];
OCTData[id] = OCTData[changeID];
OCTData[changeID] = value;
}