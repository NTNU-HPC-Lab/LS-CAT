#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ConnectPointsStatus(int* PointType_BestN, int* ConnectStatus, int size, int rows, int ChooseBestN, int ConnectRadius)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= size * rows * ChooseBestN)						// 判斷是否超出大小
return;

// 算 Index
int sizeIndex = id / (rows * ChooseBestN);
int tempID = id % (rows * ChooseBestN);
int rowIndex = tempID / ChooseBestN;
int chooseIndex = tempID % ChooseBestN;

// 代表這個點沒有有效的點
if (PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] == -1)
return;

// 如果是有效的點，就繼續往下追
int finalPos = min(rowIndex + ConnectRadius, rows);		// 截止條件
for (int i = rowIndex + 1; i < finalPos; i++)
{
for (int j = 0; j < ChooseBestN; j++)
{
// 下一個點的位置 (第 i 個 row 的點)
// 然後的第 1 個點
if (PointType_BestN[sizeIndex * rows * ChooseBestN + i * ChooseBestN + j] != -1)
{
// 前面項為現在這個點
// 後面項為往下的點
int diffX = PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] -
PointType_BestN[sizeIndex * rows * ChooseBestN + i * ChooseBestN + j];
int diffY = i - rowIndex;
int Radius = diffX * diffX + diffY * diffY;

// 0 沒有用到喔
if (Radius < ConnectRadius * ConnectRadius)
{
// 張數的位移 + Row 的位移 + 現在在 Top N 的點 + 半徑的位移 + 往下 Top N 的結果
int index = sizeIndex * rows * ChooseBestN * ConnectRadius * ChooseBestN +			// 張數
rowIndex * ChooseBestN * ConnectRadius * ChooseBestN +					// Row
chooseIndex * ConnectRadius * ChooseBestN +								// 現在在 Top N
(i - rowIndex) * ChooseBestN +											// 半徑
j;
ConnectStatus[index] = Radius;
}
}
}
}
}