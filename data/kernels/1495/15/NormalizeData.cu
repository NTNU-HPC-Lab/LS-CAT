#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void NormalizeData(float* ShiftData, float MaxValue, float MinValue, int FinalDataSize)
{
// 這邊是根據資料的最大最小值，去做 Normalize 資料
int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
threadIdx.x;

// 例外判斷
if (id >= FinalDataSize)
{
printf("Normaliza Data 超出範圍\n");
return;
}

if (ShiftData[id] < MinValue)
ShiftData[id] = 0;
else if (ShiftData[id] > MaxValue)
ShiftData[id] = 1;
else
ShiftData[id] = (ShiftData[id] - MinValue) / (MaxValue - MinValue);

}