#include "hip/hip_runtime.h"
#include "includes.h"
//-----include header files, ¤Þ¤J¼ÐÀYÀÉ-----


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)			//	addKernel¨ç¼Æ
{																		//	addKernel function, addKernel¨ç¼Æ
int i = threadIdx.x;
c[i] = a[i] + b[i];
}