#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void assign_add(float *target, const float *source)
{
target[0] += source[0];
target[1] += source[1];
target[2] += source[2];
}
__global__ void	PoissonImageCloningIteration( const float *fixed, const float *mask, const float *source, float *target ,const int wt, const int ht)
{
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt*yt+xt;
const int Nt = wt*(yt-1)+xt;
const int Wt = wt*yt+xt-1;
const int St = wt*(yt+1)+xt;
const int Et = wt*yt+xt+1;
float sum[3] = {};
if(yt < ht and xt < wt){
assign_add(sum, &fixed[curt*3]);
if((yt-1) >= 0){
if(mask[Nt] > 127.0f){
assign_add(sum, &source[Nt*3]);
}
}
if((xt-1) >= 0){
if(mask[Wt] > 127.0f){
assign_add(sum, &source[Wt*3]);
}
}
if((yt+1) < ht){
if(mask[St] > 127.0f){
assign_add(sum, &source[St*3]);
}
}
if((xt+1) < wt){
if(mask[Et] > 127.0f){
assign_add(sum, &source[Et*3]);
}
}
target[curt*3+0] = sum[0]/4;
target[curt*3+1] = sum[1]/4;
target[curt*3+2] = sum[2]/4;
}
}