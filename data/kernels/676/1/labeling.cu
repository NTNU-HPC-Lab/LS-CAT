#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void labeling(const char *text, int *pos, int text_size){
int index = threadIdx.x*blockDim.y+threadIdx.y + blockDim.x*blockDim.y*(gridDim.y*blockIdx.x + blockIdx.y);
if (index >= text_size) {
return;
}
pos[index] = 0;
if (text[index] <= ' ')
return ;
for (int k = index; k >= 0; k--) {
if (text[k] <= ' ') {
pos[index] = index - k;
return;
}
}
pos[index] = index+1;

}