#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void roipool_bp_cuda_(int nProposal, int C, float *d_feats, int *proposals_offset, int *output_maxidx, float *d_output_feats){
for(int pp_id = blockIdx.x; pp_id < nProposal; pp_id += gridDim.x){
for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
int argmax_idx = output_maxidx[pp_id * C + plane];
atomicAdd(&d_feats[argmax_idx * C + plane], d_output_feats[pp_id * C + plane]);
}
}
}