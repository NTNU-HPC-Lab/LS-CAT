#include "hip/hip_runtime.h"
#include "includes.h"
/*
Ball Query with BatchIdx
Written by Li Jiang
All Rights Reserved 2020.
*/



/* ================================== ballquery_batch_p ================================== */


__global__ void ballquery_batch_p_cuda_(int n, int meanActive, float radius, const float *xyz, const int *batch_idxs, const int *batch_offsets, int *idx, int *start_len, int *cumsum) {
int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
if (pt_idx >= n) return;

start_len += (pt_idx * 2);
int idx_temp[1000];

float radius2 = radius * radius;
float o_x = xyz[pt_idx * 3 + 0];
float o_y = xyz[pt_idx * 3 + 1];
float o_z = xyz[pt_idx * 3 + 2];

int batch_idx = batch_idxs[pt_idx];
int start = batch_offsets[batch_idx];
int end = batch_offsets[batch_idx + 1];

int cnt = 0;
for(int k = start; k < end; k++){
float x = xyz[k * 3 + 0];
float y = xyz[k * 3 + 1];
float z = xyz[k * 3 + 2];
float d2 = (o_x - x) * (o_x - x) + (o_y - y) * (o_y - y) + (o_z - z) * (o_z - z);
if(d2 < radius2){
if(cnt < 1000){
idx_temp[cnt] = k;
}
else{
break;
}
++cnt;
}
}

start_len[0] = atomicAdd(cumsum, cnt);
start_len[1] = cnt;

int thre = n * meanActive;
if(start_len[0] >= thre) return;

idx += start_len[0];
if(start_len[0] + cnt >= thre) cnt = thre - start_len[0];

for(int k = 0; k < cnt; k++){
idx[k] = idx_temp[k];
}
}