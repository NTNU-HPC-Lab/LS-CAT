#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ballquery_batch_p_cuda_.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int n = XSIZE*YSIZE;
int meanActive = 1;
float radius = 1;
const float *xyz = NULL;
hipMalloc(&xyz, XSIZE*YSIZE);
const int *batch_idxs = NULL;
hipMalloc(&batch_idxs, XSIZE*YSIZE);
const int *batch_offsets = NULL;
hipMalloc(&batch_offsets, XSIZE*YSIZE);
int *idx = NULL;
hipMalloc(&idx, XSIZE*YSIZE);
int *start_len = NULL;
hipMalloc(&start_len, XSIZE*YSIZE);
int *cumsum = NULL;
hipMalloc(&cumsum, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ballquery_batch_p_cuda_<<<gridBlock,threadBlock>>>(n,meanActive,radius,xyz,batch_idxs,batch_offsets,idx,start_len,cumsum);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ballquery_batch_p_cuda_<<<gridBlock,threadBlock>>>(n,meanActive,radius,xyz,batch_idxs,batch_offsets,idx,start_len,cumsum);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ballquery_batch_p_cuda_<<<gridBlock,threadBlock>>>(n,meanActive,radius,xyz,batch_idxs,batch_offsets,idx,start_len,cumsum);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}