#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sec_mean_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
int start = offsets[p_id];
int end = offsets[p_id + 1];

float count = (float)(end - start);

for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
float mean = 0;
for(int i = start; i < end; i++){
mean += (inp[i * C + plane] / count);
}
out[p_id * C + plane] = mean;
}
}
}