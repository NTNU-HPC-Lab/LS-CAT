#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void roipool_fp_cuda_(int nProposal, int C, float *feats, int *proposals_offset, float *output_feats, int *output_maxidx){
for(int pp_id = blockIdx.x; pp_id < nProposal; pp_id += gridDim.x){
int start = proposals_offset[pp_id];
int end = proposals_offset[pp_id + 1];

for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
int argmax_idx = -1;
float max_val = -1e50;

for(int i = start; i < end; i++){
if(feats[i * C + plane] > max_val){
argmax_idx = i;
max_val = feats[i * C + plane];
}
}
output_maxidx[pp_id * C + plane] = argmax_idx;
output_feats[pp_id * C + plane] = max_val;
}
}
}