#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sec_max_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
int start = offsets[p_id];
int end = offsets[p_id + 1];

for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
float max_val = -1e50;
for(int i = start; i < end; i++){
if(inp[i * C + plane] > max_val){
max_val = inp[i * C + plane];
}
}
out[p_id * C + plane] = max_val;
}
}
}