#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sec_min_cuda_(int nProposal, int C, float *inp, int *offsets, float *out){
for(int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x){
int start = offsets[p_id];
int end = offsets[p_id + 1];

for(int plane = threadIdx.x; plane < C; plane += blockDim.x){
float min_val = 1e50;
for(int i = start; i < end; i++){
if(inp[i * C + plane] < min_val){
min_val = inp[i * C + plane];
}
}
out[p_id * C + plane] = min_val;
}
}
}