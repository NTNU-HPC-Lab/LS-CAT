#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateDelaysAndPhases(double * gpuDelays, double lo, double sampletime, int fftsamples, int fftchannels, int samplegranularity, float * rotationPhaseInfo, int *sampleShifts, float* fractionalSampleDelays)
{
size_t ifft = threadIdx.x + blockIdx.x * blockDim.x;
size_t iant = blockIdx.y;
int numffts = blockDim.x * gridDim.x;
double meandelay, deltadelay, netdelaysamples_f, startphase;
double d0, d1, d2, a, b;
double * interpolator = &(gpuDelays[iant*4]);
double filestartoffset = gpuDelays[iant*4+3];
float fractionaldelay;
int netdelaysamples;

// evaluate the delay for the given FFT of the given antenna

// calculate values at the beginning, middle, and end of this FFT
d0 = interpolator[0]*ifft*ifft + interpolator[1]*ifft + interpolator[2];
d1 = interpolator[0]*(ifft+0.5)*(ifft+0.5) + interpolator[1]*(ifft+0.5) + interpolator[2];
d2 = interpolator[0]*(ifft+1.0)*(ifft+1.0) + interpolator[1]*(ifft+1.0) + interpolator[2];

// use these to calculate a linear interpolator across the FFT, as well as a mean value
a = d2-d0; //this is the delay gradient across this FFT
b = d0 + (d1 - (a*0.5 + d0))/3.0; //this is the delay at the start of the FFT
meandelay = a*0.5 + b; //this is the delay in the middle of the FFT
deltadelay = a / fftsamples; // this is the change in delay per sample across this FFT window

netdelaysamples_f = (meandelay - filestartoffset) / sampletime;
netdelaysamples = __double2int_rn(netdelaysamples_f/samplegranularity) * samplegranularity;

// Save the integer number of sample shifts
sampleShifts[iant*numffts + ifft] = netdelaysamples;

// Save the fractional delay
fractionaldelay = (float)(-(netdelaysamples_f - netdelaysamples)*2*M_PI/fftsamples);  // radians per FFT channel
fractionalSampleDelays[iant*numffts + ifft] = fractionaldelay;

// set the fringe rotation phase for the first sample of a given FFT of a given antenna
startphase = b*lo;
rotationPhaseInfo[iant*numffts*2 + ifft*2] = (float)(startphase - int(startphase))*2*M_PI;
rotationPhaseInfo[iant*numffts*2 + ifft*2 + 1] = (float)(deltadelay * lo)*2*M_PI;
}