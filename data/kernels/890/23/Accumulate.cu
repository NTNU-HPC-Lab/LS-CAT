#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Accumulate(float4 *src, float4 *dest, int loop) {
const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
const size_t k = blockDim.x * gridDim.x;

dest[i] = src[i];

for (int n=1; n<loop; n++) {
dest[i].x  += src[i+n*k].x;
dest[i].y  += src[i+n*k].y;
dest[i].z  += src[i+n*k].z;
dest[i].w  += src[i+n*k].w;
}
}