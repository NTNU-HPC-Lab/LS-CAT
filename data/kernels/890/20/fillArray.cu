#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillArray(int8_t *dest, int loop) {
const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
const size_t k = blockDim.x * gridDim.x;
for (int n=0; n<loop; n++) {
dest[i+n*k] = sin((i+n*k)/(float)100.0)*30;
}
}