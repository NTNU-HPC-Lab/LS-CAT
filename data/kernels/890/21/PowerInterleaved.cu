#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PowerInterleaved(float4 *src, float4 *dest) {

const size_t i = blockDim.x * blockIdx.x + threadIdx.x;

// Cross pols
dest[i].x  += src[i].x * src[i].x + src[i].y * src[i].y;
dest[i].y  += src[i].z * src[i].z + src[i].w * src[i].w;
// Parallel pols
dest[i].z += src[i].x * src[i].z + src[i].y * src[i].w;
dest[i].w += src[i].y * src[i].z - src[i].x * src[i].w;
}