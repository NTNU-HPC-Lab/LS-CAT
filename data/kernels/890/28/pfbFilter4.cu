#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pfbFilter4(float *filtered, float *unfiltered, float *taps, const int ntaps) {

const int nfft = blockDim.x;
const int i = threadIdx.x + threadIdx.y*blockDim.x*4 + blockIdx.x*blockDim.x*blockDim.y*4;

filtered[i] = unfiltered[i] * taps[threadIdx.x];
filtered[i+nfft] = unfiltered[i+nfft] * taps[threadIdx.x];
filtered[i+nfft*2] = unfiltered[i+nfft*2] * taps[threadIdx.x];
filtered[i+nfft*3] = unfiltered[i+nfft*3] * taps[threadIdx.x];
for (int j=1; j<ntaps; j++) {
filtered[i] += unfiltered[i + j*nfft] * taps[threadIdx.x + j*nfft];
filtered[i+nfft] += unfiltered[i + (j+1)*nfft] * taps[threadIdx.x + j*nfft];
filtered[i+nfft] += unfiltered[i + (j+2)*nfft] * taps[threadIdx.x + j*nfft];
filtered[i+nfft] += unfiltered[i + (j+3)*nfft] * taps[threadIdx.x + j*nfft];
}
}