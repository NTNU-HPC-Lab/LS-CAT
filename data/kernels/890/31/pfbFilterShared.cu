#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pfbFilterShared(float *filtered, float *unfiltered, float *taps, const int ntaps) {
extern __shared__ float shared_taps[];

const int nfft = blockDim.x;
const int i = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
if (i<ntaps*nfft) {
shared_taps[i] = taps[i];
}
__syncthreads();


filtered[i] = unfiltered[i] * shared_taps[threadIdx.x];
for (int j=1; j<ntaps; j++) {
filtered[i] += unfiltered[i + j*nfft] * shared_taps[threadIdx.x + j*nfft];
}
}