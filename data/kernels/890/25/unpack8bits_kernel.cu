#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpack8bits_kernel(float *rcp, float *lcp, const int8_t *src) {

const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
const size_t j = i*2;

rcp[i] = static_cast<float>(src[j]);
lcp[i] = static_cast<float>(src[j+1]);
}