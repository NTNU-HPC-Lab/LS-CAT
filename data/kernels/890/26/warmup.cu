#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *input, float *output) {

const int i = threadIdx.x + blockIdx.x * blockDim.x;
output[i] = input[i] * input[i];
}