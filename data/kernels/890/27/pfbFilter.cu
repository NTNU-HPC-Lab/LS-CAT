#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pfbFilter(float *filtered, float *unfiltered, float *taps, const int ntaps) {

const int nfft = blockDim.x;
const int i = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x * blockDim.x * blockDim.y;

filtered[i] = unfiltered[i] * taps[threadIdx.x];
for (int j=1; j<ntaps; j++) {
filtered[i] += unfiltered[i + j*nfft] * taps[threadIdx.x + j*nfft];
}
}