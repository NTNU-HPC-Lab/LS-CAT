#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 32
#define N 2048

__global__ void matMult(float* A, float* B, float* C){
// Индекс блока
int bx = blockIdx.x;
int by = blockIdx.y;

// Индекс нити
int tx = threadIdx.x;
int ty = threadIdx.y;

float sum = 0.0;
//Индекс A[i][0]
int ia = N * BLOCK_SIZE * by + N * ty;
// Индекс B[0][j]
int ib = BLOCK_SIZE * bx + tx;


for (int k = 0; k < N; k++) {
sum += A[ia + k] * B[ib + k * N];
}
// Индекс C[i][j]
int ic = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;

//Результирующая матрица
C[ic + N * ty + tx] = sum;
}