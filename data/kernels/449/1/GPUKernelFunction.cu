#include "hip/hip_runtime.h"
#include "includes.h"
/* ==================================================================
Programmers:
Kevin Wagner
Elijah Malaby
John Casey

Omptimizing SDH histograms for input larger then global memory
==================================================================
*/



#define BOX_SIZE 23000 /* size of the data box on one dimension */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
float x_pos;
float y_pos;
float z_pos;
} atom;

unsigned long long * histogram;		/* list of all buckets in the histogram */
unsigned long long  PDH_acnt;	/* total number of data points */
int block_size;		/* Number of threads per block */
int num_buckets;	/* total number of buckets in the histogram */
float   PDH_res;	/* value of w */
atom * atom_list;	/* list of all data points */
unsigned long long * histogram_GPU;
unsigned long long * temp_histogram_GPU;
atom * atom_list_GPU;


__device__ void block_to_block (atom * block_a, atom * block_b, int b_length, unsigned long long * histogram, float resolution) {
atom me = block_a[threadIdx.x];
for(int i = 0; i < b_length; i++)
atomicAdd(&(histogram[(int)(sqrt((me.x_pos - block_b[i].x_pos) * (me.x_pos - block_b[i].x_pos) +
(me.y_pos - block_b[i].y_pos) * (me.y_pos - block_b[i].y_pos) +
(me.z_pos - block_b[i].z_pos) * (me.z_pos - block_b[i].z_pos)) / resolution)]),
1);
}
__global__ void GPUKernelFunction (unsigned long long PDH_acnt, float PDH_res, atom * atom_list_GPU, unsigned long long * histogram_GPU, int num_buckets) {

extern __shared__ unsigned long long SHist[];
/* assign register values */
int i, h_pos;
float dist;
atom * my_block = &atom_list_GPU[blockIdx.x * blockDim.x];
atom temp_atom_1 = my_block[threadIdx.x];

for(h_pos=threadIdx.x; h_pos < num_buckets; h_pos+=blockDim.x)
SHist[h_pos] = 0;

__syncthreads();

/* loop through all points in atom list calculating distance from current point to all further points */
for (i = threadIdx.x + 1; i < blockDim.x && i+blockIdx.x*blockDim.x < PDH_acnt; i++)
{
atom temp_atom_2 = my_block[i];
dist = sqrt((temp_atom_1.x_pos - temp_atom_2.x_pos) * (temp_atom_1.x_pos - temp_atom_2.x_pos) +
(temp_atom_1.y_pos - temp_atom_2.y_pos) * (temp_atom_1.y_pos - temp_atom_2.y_pos) +
(temp_atom_1.z_pos - temp_atom_2.z_pos) * (temp_atom_1.z_pos - temp_atom_2.z_pos));
h_pos = (int)(dist / PDH_res);
atomicAdd(&(SHist[h_pos]), 1);
}
__syncthreads();
for(i=blockIdx.x+1; i < gridDim.x-1; i++)
block_to_block(my_block,
&atom_list_GPU[i*blockDim.x],
blockDim.x,
SHist,
PDH_res);
block_to_block(my_block,
&atom_list_GPU[i*blockDim.x],
PDH_acnt-i*blockDim.x, // Last block may be small
SHist,
PDH_res);
__syncthreads();
for(h_pos = threadIdx.x; h_pos < num_buckets; h_pos += blockDim.x)
*(histogram_GPU+(num_buckets*blockIdx.x)+h_pos) += SHist[h_pos];
}