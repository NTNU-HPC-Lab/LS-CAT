#include "hip/hip_runtime.h"
#include "includes.h"
/* ==================================================================
Programmers:
Kevin Wagner
Elijah Malaby
John Casey

Omptimizing SDH histograms for input larger then global memory
==================================================================
*/



#define BOX_SIZE 23000 /* size of the data box on one dimension */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
float x_pos;
float y_pos;
float z_pos;
} atom;

unsigned long long * histogram;		/* list of all buckets in the histogram */
unsigned long long  PDH_acnt;	/* total number of data points */
int block_size;		/* Number of threads per block */
int num_buckets;	/* total number of buckets in the histogram */
float   PDH_res;	/* value of w */
atom * atom_list;	/* list of all data points */
unsigned long long * histogram_GPU;
unsigned long long * temp_histogram_GPU;
atom * atom_list_GPU;


__global__ void kernelSumHistogram( unsigned long long int *InputHists, unsigned long long int *hist, int num_atoms, int num_buckets, int block_size) {
unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
int h_pos = tid;
unsigned long long int NumberOfSumLoop = 0;
NumberOfSumLoop = (num_atoms)/block_size + ((num_atoms%block_size) ? 1:0);

while(h_pos < num_buckets) {
unsigned long long int tmpAns = 0;
for(int i=0;i<NumberOfSumLoop;i++){
tmpAns = tmpAns + *(InputHists+(i*num_buckets)+h_pos);
}
hist[h_pos] = tmpAns;
h_pos += blockDim.x * gridDim.x;
}
__syncthreads();
}