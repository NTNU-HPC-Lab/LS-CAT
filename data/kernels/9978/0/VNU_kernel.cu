#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _VNU_KERNEL_H_
#define _VNU_KERNEL_H_


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)



/*_________________________________________________Kernel_____________________________________________________*/



#endif // #ifndef _VNU_KERNEL_H_



/*_____________________________________________Begin CN Kernel___________________________________________________*/
#ifndef _CNU_KERNEL_H_
#define _CNU_KERNEL_H_


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _CNU_KERNEL_H_
__global__ void VNU_kernel(short int* device_array, short int* offset_array, short int* sign_array, short int* results_array)
{


/*_________________________________________Shared Memory Allocation____________________________________________*/

__shared__ short int  offset;                     // Memory offset values to be read from global memory
__shared__ short int thread_Id;
__shared__ short int current_Index;
/*_____________________________________Get access to thread ID and Block ID____________________________________*/

// access current thread id
thread_Id = threadIdx.x;

// Index for global memory
current_Index = ((blockIdx.x * blockDim.x + thread_Id)*2);


/*__________________________Each Thread gets its global memory variables and index(offset)_____________________*/

// Get offsets from global memory... currently these are set to zero for simplicity
offset = offset_array[current_Index];


/*_______________________________________________Begin VN_______________________________________________________*/



short int sign = 0;
short int input1 = results_array[current_Index + offset];
short int input2 = results_array[current_Index + offset +(1)];
short int input3 = device_array[(current_Index/2) + offset];

short int sum = (input1 + input2 + input3);

short int output1 = (sum - input1);
short int output2 = (sum - input2);

if(sum < 0){
sign = 1;
}

/*_________________________________Record Results back to Device Memory________________________________________*/
//Write outputs to the same addresses read initially from the global memory to get the input integers

results_array[current_Index + offset] = output1;
results_array[current_Index + offset +(1)] = output2;
sign_array[current_Index + offset] = sign;
sign_array[current_Index + offset +(1)] = sign;
}