#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _VNU_KERNEL_H_
#define _VNU_KERNEL_H_


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)



/*_________________________________________________Kernel_____________________________________________________*/



#endif // #ifndef _VNU_KERNEL_H_



/*_____________________________________________Begin CN Kernel___________________________________________________*/
#ifndef _CNU_KERNEL_H_
#define _CNU_KERNEL_H_


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _CNU_KERNEL_H_
__global__ void CNU_kernel(short int* offset_array, short int* results_array)
{

/*_________________________________________Shared Memory Allocation____________________________________________*/

__shared__ short int  offset;                     // Memory offset values to be read from global memory
__shared__ short int thread_Id;
__shared__ short int current_Index;
/*_____________________________________Get access to thread ID and Block ID____________________________________*/

// access current thread id
thread_Id = threadIdx.x;

// Index for global memory
current_Index = ((blockIdx.x * blockDim.x + thread_Id)*3);


/*__________________________Each Thread gets its global memory variables and index(offset)_____________________*/

// Get offsets from global memory... currently these are set to zero for simplicity
offset = offset_array[current_Index];




/*___________________________________________CN Kernel Logic______________________________________________________*/

short int input1 = results_array[current_Index + offset];
short int input2 = results_array[current_Index + offset +(1)];
short int input3 = results_array[current_Index + offset +(2)];

short int min1 = 0;
short int min2 = 0;
short int agr = 1; //aggregate sign

if(input1 < 0){
agr = agr*(-1);
}
if(input2 < 0){
agr = agr*(-1);
}
if(input3 < 0){
agr = agr*(-1);
}

//Check first two inputs to get initial min1 and min2
if(abs(input1) <= abs(input2)){
min1 = input1;
min2 = input2;
}
else{
min1 = input2;
min2 = input1;
}

//Check input3 against min1 and min2
if(abs(input3) <= abs(min1)){
min2 = min1;
min1 = input3;
}
else
if(abs(input3) <= abs(min2)){
min2 = input3;
}


/*_________________________________Record Results back to Device Memory________________________________________*/
//Write outputs to the same addresses read initially from the global memory to get the input integers

results_array[current_Index + offset] = min2*agr;
results_array[current_Index + offset +(1)] = min1*agr;
results_array[current_Index + offset +(2)] = min1*agr;

}