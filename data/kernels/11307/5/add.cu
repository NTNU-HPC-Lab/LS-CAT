#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
while (tid < N)
{
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}
}