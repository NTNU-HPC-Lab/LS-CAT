#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
int tid = blockIdx.x; // handle the data at this index
if (tid < N)
c[tid] = a[tid] + b[tid];
}