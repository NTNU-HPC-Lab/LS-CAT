#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floyd2DKernel(int * M, const int nverts, const int k){
int jj = blockIdx.x * blockDim.x + threadIdx.x; // indice filas
int ii = blockIdx.y * blockDim.y + threadIdx.y; // indice columnas
int tid = (ii * nverts) + jj;
int i = tid/nverts;
int j = tid - i * nverts;
//printf ("Fila %u, Columna %u => Thread id %d.\n", i, j, tid);

if(i < nverts && j < nverts){
if (i!=j && i!=k && j!=k) {
int ik = (i*nverts) + k;
int kj = (k*nverts) + j;
int ij = (i*nverts) + j;
int aux = M[ik]+M[kj];

int vikj = min(aux, M[ij]);
M[ij] = vikj;
}
}
}