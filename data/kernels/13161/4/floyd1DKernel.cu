#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floyd1DKernel(int * M, const int nverts, const int k){
int ii = blockIdx.x * blockDim.x + threadIdx.x;    // indice filas, coincide con ij
int i = ii/nverts;
int j = ii - i * nverts;

if(i < nverts && j < nverts){
int kj = (k*nverts) + j;
// printf("TID = %u \n\tI = %u => \tM[%u] = %u \n \tK = %u => \tM[%u] = %u  \n", ii, i, ii, M[ii], k, kj, M[kj]);
if (i!=j && i!=k && j!=k) {
int ik = (i*nverts) + k;
// int kj = (k*nverts) + j;
M[ii] = min(M[ik] + M[kj], M[ii]);
}
}
}