#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matAdd(float *A, float *B, float *C, int N){
// Las matrices se recorren con la ordenacion de Fortran
int j = blockIdx.x * blockDim.x + threadIdx.x; // indice filas
int i = blockIdx.y * blockDim.y + threadIdx.y; // indice columnas
int tid = (i * N) + j;

if(i < N && j < N)
C[tid] = A[tid] + B[tid];
}