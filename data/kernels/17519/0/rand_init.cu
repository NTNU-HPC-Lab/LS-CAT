#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE  1
// returns random integer from 1 to lim
__global__ void rand_init(long *a,long seed)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
//long a = 100001;
a[tid] = seed + tid;
}