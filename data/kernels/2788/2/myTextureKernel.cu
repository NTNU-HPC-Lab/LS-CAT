#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myTextureKernel(hipSurfaceObject_t SurfObj, size_t width, size_t height)
{
for (int idy = blockIdx.y * blockDim.y + threadIdx.y;
idy < height;
idy += blockDim.y * gridDim.y)
{
for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx < width;
idx += blockDim.x * gridDim.x)
{
uchar4 data = make_uchar4(255,255,255,255);
// Read from input surface
//surf2Dread(&data,  inputSurfObj, x * sizeof(uchar4), y);
// Write to output surface
surf2Dwrite(data, SurfObj, idx * sizeof(uchar4), idy);
}
}
}