#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
__shared__ int local_mem[256];
int iter, i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int local_tid = threadIdx.x;
int local_dim = blockDim.x;
int minThreadInThisBlock = blockIdx.x * blockDim.x;
int maxThreadInThisBlock = minThreadInThisBlock + (blockDim.x - 1);

if (maxThreadInThisBlock >= N)
{
local_dim = N - minThreadInThisBlock;
}

for (iter = 0; iter < iters; iter++)
{
if (tid < N)
{
local_mem[local_tid] = input_values[tid];
}

// Required for correctness
// __syncthreads();

/*
* Perform the local reduction across values written to shared memory
* by threads in this thread block.
*/
if (local_tid == 0)
{
int sum = 0;

for (i = 0; i < local_dim; i++)
{
sum = sum + local_mem[i];
}

atomicAdd(shared_var, sum);
}

// Required for correctness
// __syncthreads();
}
}