#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(int *out, int N) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < N)
{
out[tid] = 0;
}
}