#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 4;

// unrolling 4
if (idx + 3 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4;
}

__syncthreads();

// in-place reduction in global memory
if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vsmem = idata;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}