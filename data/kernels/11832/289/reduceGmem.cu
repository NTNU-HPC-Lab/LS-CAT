#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= n) return;

// in-place reduction in global memory
if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vsmem = idata;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}