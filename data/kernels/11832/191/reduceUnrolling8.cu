#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling8 (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8
if (idx + 7 * blockDim.x < n)
{
// g_idata[idx] =
//     g_idata[idx]+
//     g_idata[idx+blockDim.x] +
//     g_idata[idx+2*blockDim.x] +
//     g_idata[idx+3*blockDim.x] +
//     g_idata[idx+4*blockDim.x] +
//     g_idata[idx+5*blockDim.x] +
//     g_idata[idx+6*blockDim.x] +
//     g_idata[idx+7*blockDim.x];

int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
int b1 = g_idata[idx + 4 * blockDim.x];
int b2 = g_idata[idx + 5 * blockDim.x];
int b3 = g_idata[idx + 6 * blockDim.x];
int b4 = g_idata[idx + 7 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}