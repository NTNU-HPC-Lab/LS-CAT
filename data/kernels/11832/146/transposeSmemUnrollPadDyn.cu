#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemUnrollPadDyn (float *out, float *in, const int nx, const int ny)
{
// dynamic shared memory
extern __shared__ float tile[];

unsigned int ix = blockDim.x * blockIdx.x * 2 + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

unsigned int ti = iy * nx + ix;

unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
unsigned int irow = bidx / blockDim.y;
unsigned int icol = bidx % blockDim.y;

// coordinate in transposed matrix
unsigned int ix2 = blockDim.y * blockIdx.y + icol;
unsigned int iy2 = blockDim.x * 2 * blockIdx.x + irow;
unsigned int to = iy2 * ny + ix2;

// transpose with boundary test
if (ix + blockDim.x < nx && iy < ny)
{
// load data from global memory to shared memory
unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) +
threadIdx.x;
tile[row_idx]       = in[ti];
tile[row_idx + BDIMX] = in[ti + BDIMX];

// thread synchronization
__syncthreads();

unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
out[to] = tile[col_idx];
out[to + ny * BDIMX] = tile[col_idx + BDIMX];
}
}