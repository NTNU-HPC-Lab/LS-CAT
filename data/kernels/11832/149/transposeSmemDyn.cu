#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemDyn(float *out, float *in, int nx, int ny)
{
// dynamic shared memory
extern __shared__ float tile[];

// coordinate in original matrix
unsigned int  ix, iy, ti, to;
ix = blockDim.x * blockIdx.x + threadIdx.x;
iy = blockDim.y * blockIdx.y + threadIdx.y;

// linear global memory index for original matrix
ti = iy * nx + ix;

// thread index in transposed block
unsigned int row_idx, col_idx, irow, icol;
row_idx = threadIdx.y * blockDim.x + threadIdx.x;
irow    = row_idx / blockDim.y;
icol    = row_idx % blockDim.y;
col_idx = icol * blockDim.x + irow;

// coordinate in transposed matrix
ix = blockDim.y * blockIdx.y + icol;
iy = blockDim.x * blockIdx.x + irow;

// linear global memory index for transposed matrix
to = iy * ny + ix;

// transpose with boundary test
if (ix < nx && iy < ny)
{
// load data from global memory to shared memory
tile[row_idx] = in[ti];

// thread synchronization
__syncthreads();

// store data to global memory from shared memory
out[to] = tile[col_idx];
}
}