#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPad(float *out, float *in, int nx, int ny)
{
// static shared memory with padding
__shared__ float tile[BDIMY][BDIMX + IPAD];

// coordinate in original matrix
unsigned int  ix, iy, ti, to;
ix = blockDim.x * blockIdx.x + threadIdx.x;
iy = blockDim.y * blockIdx.y + threadIdx.y;

// linear global memory index for original matrix
ti = iy * nx + ix;

// thread index in transposed block
unsigned int bidx, irow, icol;
bidx = threadIdx.y * blockDim.x + threadIdx.x;
irow = bidx / blockDim.y;
icol = bidx % blockDim.y;

// coordinate in transposed matrix
ix = blockDim.y * blockIdx.y + icol;
iy = blockDim.x * blockIdx.x + irow;

// linear global memory index for transposed matrix
to = iy * ny + ix;

// transpose with boundary test
if (ix < nx && iy < ny)
{
// load data from global memory to shared memory
tile[threadIdx.y][threadIdx.x] = in[ti];

// thread synchronization
__syncthreads();

// store data to global memory from shared memory
out[to] = tile[icol][irow];
}
}