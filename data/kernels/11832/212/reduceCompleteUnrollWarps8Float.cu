#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8Float (float *g_idata, float *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

// convert global data pointer to the local pointer of this block
float *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8
if (idx + 7 * blockDim.x < n)
{
float a1 = g_idata[idx];
float a2 = g_idata[idx + blockDim.x];
float a3 = g_idata[idx + 2 * blockDim.x];
float a4 = g_idata[idx + 3 * blockDim.x];
float b1 = g_idata[idx + 4 * blockDim.x];
float b2 = g_idata[idx + 5 * blockDim.x];
float b3 = g_idata[idx + 6 * blockDim.x];
float b4 = g_idata[idx + 7 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}

__syncthreads();

// in-place reduction and complete unroll
if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile float *vsmem = idata;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}