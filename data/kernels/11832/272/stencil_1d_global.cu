#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d_global (float* in, float* out, float * dcoef)
{
// shared memory
__shared__ float smem[BDIM + 2 * RADIUS];

// index to global memory
int idx = threadIdx.x + blockIdx.x * blockDim.x;

// index to shared memory for stencil calculatioin
int sidx = threadIdx.x + RADIUS;

// Read data from global memory into shared memory
smem[sidx] = in[idx];

// read halo part to shared memory
if (threadIdx.x < RADIUS)
{
smem[sidx - RADIUS] = in[idx - RADIUS];
smem[sidx + BDIM] = in[idx + BDIM];
}

// Synchronize (ensure all the data is available)
__syncthreads();

// Apply the stencil
float tmp = 0.0f;
#pragma unroll

for (int i = 1; i <= RADIUS; i++)
{
tmp += dcoef[i] * (smem[sidx + i] - smem[sidx - i]);
}

// Store the result
out[idx] = tmp;
}