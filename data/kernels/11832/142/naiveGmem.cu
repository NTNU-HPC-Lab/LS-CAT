#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naiveGmem(float *out, float *in, const int nx, const int ny)
{
// matrix coordinate (ix,iy)
unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

// transpose with boundary test
if (ix < nx && iy < ny)
{
out[ix * ny + iy] = in[iy * nx + ix];
}
}