#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeRow(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
unsigned int row = iy * gridDim.x * blockDim.x + ix;

if (row < ny)
{
int row_start = row * nx;
int row_end = (row + 1) * nx;
int col_index = row;
for (int i = row_start; i < row_end; i++) {
out[col_index] = in[i];
col_index += nx;
}
}
}