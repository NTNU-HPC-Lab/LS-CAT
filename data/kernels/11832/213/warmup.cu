#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

if (ix < nx && iy < ny)
{
out[iy * nx + ix] = in[iy * nx + ix];
}
}