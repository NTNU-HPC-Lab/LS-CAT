#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void standard_kernel(float a, float *out, int iters)
{
int i;
int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

if(tid == 0)
{
float tmp;

for (i = 0; i < iters; i++)
{
tmp = powf(a, 2.0f);
}

*out = tmp;
}
}