#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int  n)
{
__shared__ int smem[DIM];

// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check
if (idx >= n) return;

smem[tid] = idata[tid];
__syncthreads();

// in-place reduction in global memory
for (int stride = 1; stride < blockDim.x; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
smem[tid] += smem[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = smem[0];
}