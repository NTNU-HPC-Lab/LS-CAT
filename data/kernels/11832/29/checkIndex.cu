#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex(void)
{
printf("threadIdx:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);
printf("blockIdx:(%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);

printf("blockDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
printf("gridDim:(%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);

}