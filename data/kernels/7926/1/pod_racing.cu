#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pod_racing(unsigned int *d_rand, unsigned int *win, unsigned int *loss, unsigned int size, int *iter) {
int index = threadIdx.x + blockDim.x*blockIdx.x;
const unsigned int flips[] = { 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 0, 1 };
if (index < size) {
//printf("%d ", iter[0]);
if ((d_rand[index] % 2) != flips[iter[0]]) {
iter[0] = 0;
loss[index] = 1;
//printf("loss ");
}
else {
iter[0] = iter[0] + 1;
if (iter[0] == 15) {
win[index] = 1;
iter[0] = 0;
//printf("win ");
}
}
}
}