#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_arrays_gpu(unsigned int * a, unsigned int * b, int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x;

if (index < size) {
a[0] = a[0] + b[index];
//printf("%u ", a[0]);
}
}