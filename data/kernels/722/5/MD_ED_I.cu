#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MD_ED_I(float *S, float *T, int trainSize, int window_size, int dimensions, float *data_out, int task, int gm) {

int idx, offset_x;
float sumErr = 0;
long long int i, j;


if(gm == 0){

extern __shared__ float sh_mem[];

float *T2 = (float *)sh_mem;
float *DTW_single_dim =
(float *)&sh_mem[dimensions * window_size]; // offset on the shared memory
// for the segment T2

if (task == 0) {
idx = threadIdx.x * dimensions + threadIdx.y;
offset_x = ((blockDim.x * blockDim.y * window_size) * blockIdx.x) +
idx * window_size;

if (((blockDim.x * blockDim.y * blockIdx.x) + idx) >=
trainSize * dimensions) // 120=train_size
return;

} else { // SUBSEQ_SEARCH

idx = threadIdx.x * dimensions + threadIdx.y;
offset_x =
(blockDim.x * blockIdx.x) +
((threadIdx.y * trainSize) +
threadIdx.x); // use blockIdx and other measure to set well the offset

if ((idx + window_size) > trainSize)
return;
}

if (idx == 0) {
for (i = 0; i < dimensions; i++)
for (j = 0; j < window_size; j++)
*(T2 + (window_size * i + j)) = T[window_size * i + j];
}
__syncthreads();

for (j = 0; j < window_size; j++)
sumErr += (S[offset_x + j] - T2[window_size * threadIdx.y + j]) *
(S[offset_x + j] - T2[window_size * threadIdx.y + j]);

DTW_single_dim[idx] = sqrt(sumErr);

__syncthreads();

if (idx == 0) {
for (i = 0; i < blockDim.x; i++) {
data_out[(blockIdx.x * blockDim.x) + i] = 0.0;
for (j = 0; j < blockDim.y; j++) {
data_out[(blockIdx.x * blockDim.x) + i] +=
DTW_single_dim[i * dimensions + j]; // rivedere formula!
}
}
}

}
else {

extern __shared__ float DTW_single_dim[];

if (task == 0) {
idx = threadIdx.x * dimensions + threadIdx.y;
offset_x = ((blockDim.x * blockDim.y * window_size) * blockIdx.x) +
idx * window_size;

if (((blockDim.x * blockDim.y * blockIdx.x) + idx) >=
trainSize * dimensions) // 120=train_size
return;

} else { // SUBSEQ_SEARCH

idx = threadIdx.x * dimensions + threadIdx.y;
offset_x =
(blockDim.x * blockIdx.x) +
((threadIdx.y * trainSize) +
threadIdx.x); // use blockIdx and other measure to set well the offset

if ((idx + window_size) > trainSize)
return;
}

// if (idx == 0) {
//   for (i = 0; i < dimensions; i++)
//     for (j = 0; j < window_size; j++)
//       *(T2 + (window_size * i + j)) = T[window_size * i + j];
// }
// __syncthreads();

for (j = 0; j < window_size; j++)
sumErr += (S[offset_x + j] - T[window_size * threadIdx.y + j]) *
(S[offset_x + j] - T[window_size * threadIdx.y + j]);

DTW_single_dim[idx] = sqrt(sumErr);

__syncthreads();

if (idx == 0) {
for (i = 0; i < blockDim.x; i++) {
data_out[(blockIdx.x * blockDim.x) + i] = 0.0;
for (j = 0; j < blockDim.y; j++) {
data_out[(blockIdx.x * blockDim.x) + i] +=
DTW_single_dim[i * dimensions + j]; // rivedere formula!
}
}
}

}

}