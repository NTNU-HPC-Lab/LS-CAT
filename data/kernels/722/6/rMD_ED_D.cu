#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rMD_ED_D(float *S, float *T, int window_size, int dimensions, float *data_out, int trainSize, int gm) {

long long int i, j, p;
float sumErr = 0, dd = 0;
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (gm == 0) {

extern __shared__ float T2[];

// offset training set
int s = dimensions * 2 * window_size * (idx / window_size);
int t = s + idx % window_size;

if (idx >= (trainSize * window_size)) //
return;

if (threadIdx.x == 0) {
for (i = 0; i < dimensions; i++)
for (j = 0; j < window_size; j++)
T2[window_size * i + j] = T[window_size * i + j];
}
__syncthreads();

for (j = 0; j < window_size; j++) {
dd = 0;
for (p = 0; p < dimensions; p++)
dd += (S[(t + p * 2 * window_size) + j] - T2[(p * window_size) + j]) *
(S[(t + p * 2 * window_size) + j] - T2[(p * window_size) + j]);

sumErr += dd;
}
data_out[idx] = sqrt(sumErr);
} else {

int s = dimensions * 2 * window_size * (idx / window_size);
int t = s + idx % window_size;

if (idx >= (trainSize * window_size))
return;

for (j = 0; j < window_size; j++) {
dd = 0;
for (p = 0; p < dimensions; p++)
dd += (S[(t + p * 2 * window_size) + j] - T[(p * window_size) + j]) *
(S[(t + p * 2 * window_size) + j] - T[(p * window_size) + j]);

sumErr += dd;
}
data_out[idx] = sqrt(sumErr);
}
}