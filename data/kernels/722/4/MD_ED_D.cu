#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MD_ED_D(float *S, float *T, int trainSize, int window_size, int dimensions, float *data_out, int task, int gm) {

long long int i, j, p;
float sumErr = 0, dd = 0;
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (gm == 0) {
extern __shared__ float T2[];

int t, offset;
if (task == 0) {

offset = window_size;

int wind = dimensions * window_size;
t = idx * wind;
if ((idx * wind) + wind >
trainSize * wind) // CHANGE FORMULA 120=train_size
return;

if (threadIdx.x == 0) {
for (i = 0; i < dimensions; i++)
for (j = 0; j < window_size; j++)
T2[window_size * i + j] = T[window_size * i + j];
}

__syncthreads();
} else {

// in this case 'trainSize' is the number of subsequence to search 'nss',
// that is, the length of dataset to perform on
offset = trainSize;

t = idx;
if ((idx + window_size) > trainSize)
return;

if (threadIdx.x == 0) {
for (i = 0; i < dimensions; i++)
for (j = 0; j < window_size; j++)
T2[window_size * i + j] = T[window_size * i + j];
}
__syncthreads();
}

for (j = 0; j < window_size; j++) {
dd = 0;
for (p = 0; p < dimensions; p++)
dd += (S[(t + p * offset) + j] - T2[(p * window_size) + j]) *
(S[(t + p * offset) + j] - T2[(p * window_size) + j]);

sumErr += dd;
}
data_out[idx] = sqrt(sumErr);
} else {

int t, offset;
if (task == 0) {

offset = window_size;

int wind = dimensions * window_size;
t = idx * wind;
if ((idx * wind) + wind > trainSize * wind)
return;
} else {

// in this case 'trainSize' is the number of subsequence to search 'nss',
// that is, the length of dataset to perform on
offset = trainSize;

t = idx;
if ((idx + window_size) > trainSize)
return;
}

for (j = 0; j < window_size; j++) {
dd = 0;
for (p = 0; p < dimensions; p++)
dd += (S[(t + p * offset) + j] - T[(p * window_size) + j]) *
(S[(t + p * offset) + j] - T[(p * window_size) + j]);

sumErr += dd;
}
data_out[idx] = sqrt(sumErr);
}
}