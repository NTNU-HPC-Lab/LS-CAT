#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clear_best_distances(int *best_distances, int rays_number) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i >= rays_number)
return;

best_distances[i] = INT32_MAX;
}