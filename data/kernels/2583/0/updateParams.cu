#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void updateParams(int N, int M, float alpha, float beta1, float beta2, float t, float *PARAMS, float *GRADS, float *m, float *v)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float beta1r = __fsub_rn(1.0, beta1);
float alphar = __fmul_rn(-alpha, __frcp_rn(__fsub_rn(1.0, __powf(beta1, t))));

if (i < N && j < M)
{
m[index] = __fmaf_rn(beta1, m[index], __fmul_rn(beta1r, GRADS[index]));
v[index] = fmaxf(fmaxf(__fmul_rn(beta2, v[index]), fabsf(GRADS[index])), 1.0e-16);
PARAMS[index] = __fmaf_rn(alphar,__fdividef(m[index], v[index]), PARAMS[index]);


//m[index] = beta1*m[index] + (1 - beta1)*GRADS[index];

//float a = beta2*v[index];
// float b = ((GRADS[index])>(0))?(GRADS[index]):(-GRADS[index]);
//float c = fmaxf(a, fabsf(GRADS[index])); // ((a)>(fabsf(GRADS[index]))?(a):(b);
//v[index] = fmaxf(c, 1.0e-16); // ((c)>(1.0e-16))?(c):(1.0e-16);
//float tmp = alpha/(1.0-powf(beta1, t));
//PARAMS[index] = PARAMS[index] - (alpha/(1.0-__powf(beta1, t)))*m[index]/v[index];
//PARAMS[index] = tmp*m[index]/v[index];
}
}