#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void elMul(int N, int M, float *X1, float *X2)
{


int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
X1[index] = __fmul_rn(X1[index], X2[index]);
}
}