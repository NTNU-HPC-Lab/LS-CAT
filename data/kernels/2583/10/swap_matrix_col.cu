#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void swap_matrix_col(int N, int C, float *X, float *V)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int index = (C-1)*N + i;

if (i < N)
{
float a = X[index];
X[index] = V[i];
V[i] = a;
}
}