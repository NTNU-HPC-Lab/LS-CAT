#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void normLogErr(int N, int M, float *A, float *Y)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;
int L = N*M;

if (i < N && j < M)
{
// A2 in this case is stored in the doubled rows of A, the length of A is
// doublt that of Y
float a = __expf(__fmul_rn(2.0, A[index+L]));
A[index] = __fmul_rn(a, __fmaf_rn(0.5, __fmul_rn(Y[index], Y[index]), __fsub_rn(__fmul_rn(0.5, __fmul_rn(A[index], A[index])),  __fmul_rn(A[index], Y[index]))));
A[index+L] = __fsub_rn(0.9189385332, A[index+L]); // stick final sum factor in 2nd part of A so when it sums to total the cost will be correct
// A[index] = a*(A[index]*(0.5*A[index] - Y[index]) + 0.5*Y[index]*Y[index]);
// A[index+L] = __fsub_rn(0.9189385332, A[index+L]);
}
}