#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void elSq(int N, int M, float *Mat)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
Mat[index] = __fmul_rn(Mat[index], Mat[index]);
}
}