#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void absErr(int N, int M, float *A, float *Y)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
A[index] = fabsf(__fsub_rn(A[index], Y[index]));
// A[index] = abs(A[index]-Y[index])
}
}