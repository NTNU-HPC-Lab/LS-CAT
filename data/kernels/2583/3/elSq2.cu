#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void elSq2(int N, int M, float *In, float *Out)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
Out[index] = __fmul_rn(In[index], In[index]);
}
}