#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void tanhGradient(int N, int M, float *z, float *tanh_grad_z) {

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float c1 = __fdividef(2.0, 3.0);

if (i < N && j < M) {
float el = __fmul_rn(z[index], c1);
if (el > 4.97) {
z[index] = 1.7159;
tanh_grad_z[index] = 0.0;
}
else if(el < -4.97) {
z[index] = -1.7159;
tanh_grad_z[index] = 0.0;
}
else {
float x2 = __fmul_rn(el, el);
float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
float tanh = __fdividef(a, b);
z[index] = __fmul_rn(1.7159, tanh);
tanh_grad_z[index] = __fmul_rn(1.7159, __fmul_rn(__fmaf_rn(-tanh, tanh, 1.0), c1));
}
}
}