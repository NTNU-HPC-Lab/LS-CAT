#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void cauchyLogErr(int N, int M, float *A, float *Y)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;
int L = N*M;

if (i < N && j < M)
{
// A2 in this case is stored in the doubled rows of A, the length of A is
// doublt that of Y
float a = __expf(A[index+L]);
A[index] = __fmul_rn(fabsf(__fsub_rn(A[index], Y[index])), a);
A[index +L] = -__logf(__fmul_rn(0.5, a)); // stick final sum factor in 2nd part of A so when it sums to total the cost will be correct
}
}