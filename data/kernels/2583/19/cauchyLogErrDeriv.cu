#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void cauchyLogErrDeriv(int N, int M, float *A, float *Y, float *out)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;
int L = N*M;

if (i < N && j < M)
{
float a = __expf(A[index+L]);
if (A[index] > Y[index])
{
out[index] = a;
}
else if (A[index] < Y[index])
{
out[index] = -a;
}
else
{
out[index] = 0.0;
}

out[index+L] = __fmaf_rn(a, fabsf(__fsub_rn(A[index],  Y[index])), -1.0);
// A2 in this case is stored in the doubled rows of A, the length of A is
// doublt that of Y, out is the same length as A and will store both parts of the derivative
}
}