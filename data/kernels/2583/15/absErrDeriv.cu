#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void absErrDeriv(int N, int M, float *A, float *Y, float *out)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
out[index] = copysignf(1.0, __fsub_rn(A[index], Y[index]));
}
}