#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void fill_cols(int N, int M, float *X, float *V)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
X[index] = V[j];

}
}