#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void noactivationGradient(int N, int M, float *z, float *tanh_grad_z, int seed, float D) {

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float scaleFactor = __fdividef(1.0, __fsub_rn(1.0, D));

if (i < N && j < M) {
hiprandState_t state;
hiprand_init( (seed << 20) + index, 0, 0, &state);

float F = hiprand_uniform(&state);
// float F = 0.5;

if (D != 0.0) {
if (F < D) {
z[index] = 0.0;
tanh_grad_z[index] = 0.0;
}
else {
tanh_grad_z[index] = scaleFactor;
z[index] = __fmul_rn(scaleFactor, z[index]);
}
}
else {
tanh_grad_z[index] = 1.0;
}
}
}