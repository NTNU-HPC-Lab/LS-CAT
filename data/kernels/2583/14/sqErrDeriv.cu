#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void sqErrDeriv(int N, int M, float *A, float *Y, float *out)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
out[index] = __fmul_rn(2.0, __fsub_rn(A[index], Y[index]));
// Out[index] = 2*(A[index] - Y[index])
}
}