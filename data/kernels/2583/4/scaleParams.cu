#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void scaleParams(int N, int M, float c, float *Mat, float *F)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
float s = __saturatef( __fdividef(c, __fsqrt_rn(F[i])));
//float s = (c/sqrt(F[i]) < 1.0) ? c/sqrt(F[i]) : 1.0;
Mat[index] = __fmul_rn(Mat[index], s);
}
}