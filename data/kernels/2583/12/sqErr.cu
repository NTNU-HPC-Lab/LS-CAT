#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void sqErr(int N, int M, float *A, float *Y)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
float tmp = __fsub_rn(A[index], Y[index]);
A[index] = __fmul_rn(tmp, tmp);
// A[index] = (A[index]-Y[index])^2
}
}