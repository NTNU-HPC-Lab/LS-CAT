#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void tanhActivation(int N, int M, float *z)
{


int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float c1 = __fdividef(2.0, 3.0);

if (i < N && j < M)
{


float el = __fmul_rn(z[index], c1);
if (el > 4.97)
{
z[index] = 1.7159;
}
else if (el < -4.97)
{
z[index] = -1.7159;
}
else
{
float x2 = __fmul_rn(el, el);
float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
float tanh = __fdividef(a, b);
z[index] = __fmul_rn(1.7159, tanh);
}
}
}