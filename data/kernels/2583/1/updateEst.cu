#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void updateEst(int N, int M, float beta2, float scale, float *PARAMS, float *AVG, float *EST)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float beta2a = __fsub_rn(1.0, beta2);
if (i < N && j < M)
{
//AVG[index] = beta2*AVG[index] + (1.0-beta2)*PARAMS[index];
//EST[index] = scale*AVG[index];
AVG[index] = __fmaf_rn(beta2a,PARAMS[index],__fmul_rn(beta2,AVG[index]));
EST[index] = __fmul_rn(scale, AVG[index]);
}
}