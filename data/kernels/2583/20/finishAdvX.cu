#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void finishAdvX(int N, int M, float *X, float *advX)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
if (advX[index] < 0)
{
advX[index] = X[index] - 5.0e-5;
}
else if (advX[index] > 0)
{
advX[index] = X[index] + 5.0e-5;
}
else
{
advX[index] = X[index];
}

}
}