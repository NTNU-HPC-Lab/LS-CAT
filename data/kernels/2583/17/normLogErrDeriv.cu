#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void normLogErrDeriv(int N, int M, float *A, float *Y, float *out)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;
int L = N*M;

if (i < N && j < M)
{
// A2 in this case is stored in the doubled rows of A, the length of A is
// doublt that of Y, out is the same length as A and will store both parts of the derivative
float a = __expf(__fmul_rn(2.0, A[index+L]));
float b = __fsub_rn(A[index], Y[index]);
out[index] = __fmul_rn(b, a);
out[index+L] = __fsub_rn(__fmul_rn(out[index], b), 1.0);
}
}