#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void tanhGradientDropout(int N, int M, float *z, float *tanh_grad_z, int seed, float D) {

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

float c1 = __fdividef(2.0, 3.0);
float scaleFactor1 = __fdividef(1.7159, __fsub_rn(1.0, D));
float scaleFactor2 = __fdividef(-1.7159, __fsub_rn(1.0, D));

if (i < N && j < M) {
hiprandState_t state;
hiprand_init( (seed << 20) + index, 0, 0, &state);

float F = hiprand_uniform(&state);
// float F = 0.5;

if(F<D) {
z[index] = 0.0;
tanh_grad_z[index] = 0.0;
}
else {
float el = __fmul_rn(z[index], c1);
if(el > 4.97) {
z[index] = scaleFactor1;
tanh_grad_z[index] = 0.0;
}
else if(el < -4.97) {
z[index] = scaleFactor2;
tanh_grad_z[index] = 0.0;
}
else {
float x2 = __fmul_rn(el, el);
float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
float tanh = __fdividef(a, b);
z[index] = __fmul_rn(scaleFactor1, tanh);
tanh_grad_z[index] = __fmul_rn(scaleFactor1, __fmul_rn(__fmaf_rn(-tanh, tanh, 1.0), c1));
}
}
}
}