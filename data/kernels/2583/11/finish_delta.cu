#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix



extern "C"   // ensure function name to be exactly "eeTanh"
{





















}
__global__ void finish_delta(int N, int M, float *A, float *Y, float *out)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int index = j*N + i;

if (i < N && j < M)
{
out[index] = copysignf(1.0, __fsub_rn(A[index], Y[index]));

/*
if (A[index] < Y[index])
{
out[index] = -1.0;
}
else if (A[index] > Y[index])
{
out[index] = 1.0;
}
else
{
out[index] = 0.0;
}
*/

}
}