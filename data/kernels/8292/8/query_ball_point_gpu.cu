#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, const float* radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
for (int i=0;i<b;++i) {
for (int j=0;j<m;++j) {
int cnt = 0;
for (int k=0;k<n;++k) {
if (cnt == nsample)
break; // only pick the FIRST nsample points in the ball
float x2=xyz2[j*3+0];
float y2=xyz2[j*3+1];
float z2=xyz2[j*3+2];
float x1=xyz1[k*3+0];
float y1=xyz1[k*3+1];
float z1=xyz1[k*3+2];
float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
if (d<radius[0]) {
if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
for (int l=0;l<nsample;++l)
idx[j*nsample+l] = k;
}
idx[j*nsample+cnt] = k;
cnt+=1;
}
}
}
xyz1+=n*3;
xyz2+=m*3;
idx+=m*nsample;
}
}