#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_vecDouble(int *in, int *out, const int n)
{
int i = threadIdx.x;
if (i < n) {
out[i] = in[i] * 2;
}
}