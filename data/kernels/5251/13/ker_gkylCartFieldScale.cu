#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldScale(unsigned s, unsigned nv, double fact, double *out)
{
for (int n = blockIdx.x*blockDim.x + threadIdx.x + s; n < s + nv; n += blockDim.x * gridDim.x)
out[n] *= fact;
}