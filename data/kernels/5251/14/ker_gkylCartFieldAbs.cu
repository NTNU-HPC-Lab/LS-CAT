#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldAbs(unsigned s, unsigned nv, double *out)
{
for (int n = blockIdx.x*blockDim.x + threadIdx.x + s; n < s + nv; n += blockDim.x * gridDim.x)
out[n] = fabs(out[n]);
}