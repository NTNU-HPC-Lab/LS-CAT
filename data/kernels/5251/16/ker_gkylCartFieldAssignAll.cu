#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldAssignAll(unsigned s, unsigned nv, double val, double *out)
{
for (int n = blockIdx.x*blockDim.x + threadIdx.x + s; n < s + nv; n += blockDim.x * gridDim.x)
out[n] = val;
}