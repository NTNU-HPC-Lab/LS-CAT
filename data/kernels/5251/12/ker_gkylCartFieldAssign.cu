#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldAssign(unsigned s, unsigned nv, double fact, const double *inp, double *out)
{
for (int n = blockIdx.x*blockDim.x + threadIdx.x + s; n < s + nv; n += blockDim.x * gridDim.x)
out[n] = fact*inp[n];
}