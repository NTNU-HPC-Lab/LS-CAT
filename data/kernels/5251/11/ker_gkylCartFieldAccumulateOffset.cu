#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldAccumulateOffset(unsigned sInp, unsigned sOut, unsigned nCells, unsigned compStart, unsigned nCompInp, unsigned nCompOut, double fact, const double *inp, double *out) {
if (nCompInp < nCompOut) {
for (unsigned i=blockIdx.x*blockDim.x + threadIdx.x; i<nCells; i += blockDim.x * gridDim.x) {
for (unsigned c=0; c<nCompInp; ++c) {
out[sOut + i*nCompOut + compStart + c] += fact*inp[sInp + i*nCompInp + c];
}
}
}
else {
for (unsigned i=blockIdx.x*blockDim.x + threadIdx.x; i<nCells; i += blockDim.x * gridDim.x) {
for (unsigned c=0; c<nCompOut; ++c) {
out[sOut + i*nCompOut + c] += fact*inp[sInp + i*nCompInp + compStart + c];
}
}
}
}