#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_addition(const double *A, const double *B, double *C, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
C[tid] = __fadd_rd(A[tid], B[tid]);
tid += stride;
}
}