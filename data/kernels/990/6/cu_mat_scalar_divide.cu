#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_mat_scalar_divide(double *A, double B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
A[tid] = __fdiv_rd(A[tid], B);
tid += stride;
}
}