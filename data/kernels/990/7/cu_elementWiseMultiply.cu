#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_elementWiseMultiply(double *A, const double *B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
A[tid] = __fmul_rd(A[tid], B[tid]);
tid += stride;
}
}