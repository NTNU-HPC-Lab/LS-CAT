#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_mat_multiply(const double* A, const double* B, double * C, int rowsa, int colsa, int rowsb, int colsb, int rowsc, int colsc){
__shared__ double sA[32][32];   // Tile size of 32x32
__shared__ double sB[32][32];
int Row = blockDim.y*blockIdx.y + threadIdx.y;
int Col = blockDim.x*blockIdx.x + threadIdx.x;
double Cvalue = 0.0;
sA[threadIdx.y][threadIdx.x] = 0.0;
sB[threadIdx.y][threadIdx.x] = 0.0;
for (int k = 0; k < (((colsa - 1)/ 32) + 1); k++){
if ( (Row < rowsa) && (threadIdx.x + (k*32)) < colsa){
sA[threadIdx.y][threadIdx.x] = A[(Row*colsa) + threadIdx.x + (k*32)];
}
else{
sA[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();
if ( Col < colsb && (threadIdx.y + k*32) < rowsb){
sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*32)*colsb + Col];
}
else{
sB[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();

for (int j = 0; j < 32; ++j){
Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
}
__syncthreads();
}
if (Row < rowsc && Col < colsc){
C[Row*colsc + Col] = Cvalue;
}
}