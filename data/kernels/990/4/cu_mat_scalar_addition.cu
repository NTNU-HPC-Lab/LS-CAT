#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_mat_scalar_addition(double *A, const double b, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
A[tid] = __fadd_rd(A[tid], b);
tid += stride;
}
}