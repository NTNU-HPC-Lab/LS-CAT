#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_sigmoid(double* src, double* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
double tmp = __fmul_rd(src[tid], -1.0);
tmp = __expf(tmp);
tmp = __fadd_rd(tmp, 1.0);
dst[tid] = __fdividef(1.0, tmp);
tid += stride;
}
}