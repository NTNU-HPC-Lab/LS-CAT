#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_mat_scalar_multiply(double *A, double B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
A[tid] = __fmul_rd(A[tid], B);
tid += stride;
}
}