#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initancestors_noresample(int *ancestor, int np) {
int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
while (ii < np) {
ancestor[ii] = ii; //note that the next time step is the same as K time steps back. it's ok to overwrite this since we've already copied out the relevant values as a_gs
ii += BLOCKSIZE * gridDim.x;
}
}