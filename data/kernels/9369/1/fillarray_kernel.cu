#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillarray_kernel(float *x, float v, int np) {
int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
while (ii < np) {
x[ii] = v;
ii += BLOCKSIZE * gridDim.x; //grid strides
}
}