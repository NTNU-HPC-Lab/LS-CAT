#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_identity_kernel( float *a, int m, int n )
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

if( col < n && row < m)
{
a[row * n + col] = (row == col) ? 1.0f: 0.0f;
}
}