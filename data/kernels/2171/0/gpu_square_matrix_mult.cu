#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_square_matrix_mult(float *d_a, float *d_b, float *d_result, int n)
{
__shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
__shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
int tmp = 0;
int idx;

for (int sub = 0; sub < gridDim.x; ++sub)
{
idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
if(idx >= n*n)
{
// n may not divisible by BLOCK_SIZE
tile_a[threadIdx.y][threadIdx.x] = 0;
}
else
{
tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
}

idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
if(idx >= n*n)
{
tile_b[threadIdx.y][threadIdx.x] = 0;
}
else
{
tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
}
__syncthreads();

for (int k = 0; k < BLOCK_SIZE; ++k)
{
tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
}
__syncthreads();
}
if(row < n && col < n)
{
d_result[row * n + col] = tmp;
}
}