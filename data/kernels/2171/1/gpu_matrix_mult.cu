#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult(float *a,float *b, float *c, int m, int n, int k)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if( col < k && row < m)
{
for(int i = 0; i < n; i++)
{
sum += a[row * n + i] * b[i * k + col];
}
c[row * k + col] = sum;
}
}