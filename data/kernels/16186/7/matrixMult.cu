#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMult (int *a, int *b, int *c, int width)
{
int i, sum = 0;
int col = threadIdx.x + blockDim.x * blockIdx.x;
int row = threadIdx.y + blockDim.y * blockIdx.y;
if(col < width && row < width)
for (i = 0; i< width; i++)
{
sum += a[row * width + i] * b[i * width + col];
}
c[row * width + col] = sum;
}