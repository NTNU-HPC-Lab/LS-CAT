#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _cuda_add_scalar(int *in, int scalar, int n)
{
int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
while(globalIdx < n)
{
in[globalIdx] = in[globalIdx] + scalar;
globalIdx += blockDim.x * gridDim.x;
}
}