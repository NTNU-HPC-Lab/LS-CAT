#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot( int *a, int *b, int *c ) {
__shared__ int temp[THREADS_PER_BLOCK];
int index = threadIdx.x + blockIdx.x * blockDim.x;
temp[threadIdx.x] = a[index] * b[index];
__syncthreads();
if( 0 == threadIdx.x ) {
int sum = 0;
for( int i = 0; i < THREADS_PER_BLOCK; i++ )
sum += temp[i];
atomicAdd( c , sum );
}
}