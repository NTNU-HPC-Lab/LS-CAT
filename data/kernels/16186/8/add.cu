#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c ) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}