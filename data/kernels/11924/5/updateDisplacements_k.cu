#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateDisplacements_k(float4 *Ui_t, float4 *Ui_tminusdt, float *M, float4 *Ri, float4 *Fi, int maxNumForces, float4 *ABC, unsigned int numPoints)
{
int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

if (me_idx>=numPoints)
return;

float4 F = make_float4(0,0,0,0);

//	printf("Max num forces: %i\n", maxNumForces);

for (int i=0; i<maxNumForces; i++)
{
float4 force_to_add = Fi[me_idx*maxNumForces+i];
F.x += force_to_add.x;
F.y += force_to_add.y;
F.z += force_to_add.z;
}
//	printf("Accumulated node %i force: %f, %f, %f \n", me_idx, F.x, F.y, F.z);

float4 ABCi = ABC[me_idx];
float4 Uit = Ui_t[me_idx];
float4 Uitminusdt = Ui_tminusdt[me_idx];

float4 R = Ri[me_idx];
float x = ABCi.x * (R.x - F.x) + ABCi.y * Uit.x + ABCi.z * Uitminusdt.x;
float y = ABCi.x * (R.y - F.y) + ABCi.y * Uit.y + ABCi.z * Uitminusdt.y;
float z = ABCi.x * (R.z - F.z) + ABCi.y * Uit.z + ABCi.z * Uitminusdt.z;

/*	float x = ABCi.x * (-F.x) + ABCi.y * Ui_t[me_idx].x + ABCi.z * Ui_tminusdt[me_idx].x;
float y = ABCi.x * (-F.x) + ABCi.y * Ui_t[me_idx].y + ABCi.z * Ui_tminusdt[me_idx].y;
float z = ABCi.x * (-F.x ) + ABCi.y * Ui_t[me_idx].z + ABCi.z * Ui_tminusdt[me_idx].z;
*/
Ui_tminusdt[me_idx] = make_float4(x,y,z,0);//XXXXXXXXXXXXXXXXXXXXX

}