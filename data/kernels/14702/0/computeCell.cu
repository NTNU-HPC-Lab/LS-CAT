#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void applyRule(char* left, char* middle, char* right, char* res){
char a = *left;
char b = *middle;
char c = *right;

if(a == 0 && b == 0 && c == 0){
*res = 0;
}else if(a == 0 && b == 0 && c == 1){
*res = 1;
}else if(a == 0 && b == 1 && c == 0){
*res = 1;
}else if(a == 0 && b == 1 && c == 1){
*res = 1;
}else if(a == 1 && b == 0 && c == 0){
*res = 0;
}else if(a == 1 && b == 0 && c == 1){
*res = 1;
}else if(a == 1 && b == 1 && c == 0){
*res = 1;
}else if(a == 1 && b == 1 && c == 1){
*res = 0;
}
}
__global__ void computeCell(char* cellData, unsigned int* width, unsigned int* height)
{
int y = 0;
int x = 0;

/*
printf("width : %d\n", *width);
printf("height : %d\n", *height);
printf("dimblock : %d\n", blockDim.x);
printf("threadid : %d\n", threadIdx.x);
*/

for(y=1; y < (*height); ++y){
for(x=threadIdx.x; x < (*width); x += blockDim.x){
char left = ((x-1)>=0) ? cellData[(x-1) + (y-1) * (*width)] : 0;
char middle = cellData[x + (y-1) * (*width)];
char right = ((x+1) < *width) ? cellData[(x+1) + (y-1) * (*width)] : 0;

//printf("left   : (%d, %d) => %d : %d\n", x-1, y-1, (x-1) + (y-1) * (*width), left);
//printf("middle : (%d, %d) => %d : %d\n", x, y-1, (x) + (y-1) * (*width), middle);
//printf("right  : (%d, %d) => %d : %d\n", x+1, y-1, (x+1) + (y-1) * (*width), right);
applyRule(&left, &middle, &right, &cellData[x + y * (*width)]);
//printf("res    : (%d, %d) => %d : %d\n", x, y, x + y * (*width), cellData[x+y*(*width)]);

//printf("-----------\n");
}
__syncthreads();
}

/*
for(y=0; y < *height; ++y){
for(x=0; x < *width; ++x){
printf("(%d, %d) = %d\n", x, y, cellData[x+y*(*width)]);
}
}
*/
}