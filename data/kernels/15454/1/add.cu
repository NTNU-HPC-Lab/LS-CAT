#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, float a, float *x, float *y){
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n) y[i] = a*x[i] + y[i];
}