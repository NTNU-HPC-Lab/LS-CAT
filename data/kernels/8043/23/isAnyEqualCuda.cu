#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isAnyEqualCuda( const uint8_t * image, uint8_t * value, size_t valueCount, uint32_t width, uint32_t height, uint32_t * differenceCount )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if( x < width && y < height )
{
const uint32_t id = y * width + x;

bool equal = false;

for( uint32_t i = 0; i < valueCount; ++i )
{
if( image[id] == value[i] )
{
equal = true;
break;
}
}

if( equal )
atomicAdd( differenceCount, 1 );
}
}