#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
const uint32_t id = y * rowSize + x;
atomicAdd( &histogram[data[id]], 1 );
}
}