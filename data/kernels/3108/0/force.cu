#include "hip/hip_runtime.h"
#include "includes.h"

//Device Code....

__global__ void force(double *deviceq,double *devicex,double *devicey,double *devicez,double *deviceFx,double *deviceFy,double *deviceFz,double *deviceU,int N)
{
double foxij,foyij,fozij,xij,yij,zij,rij;
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j;
if(i < N-1)
{       //Anurag Dogra
for(j=i;j<N;j++)
{
if(i!=j)
{
xij = devicex[i] - devicex[j];
yij = devicey[i] - devicey[j];
zij = devicez[i] - devicez[j];

//Distance calculation
rij = sqrt((xij*xij)+(yij*yij)+(zij*zij));

foxij = foxij + ((deviceq[i]*deviceq[j]*xij)/(rij*rij*rij));
foyij = foyij + ((deviceq[i]*deviceq[j]*yij)/(rij*rij*rij));
fozij = fozij + ((deviceq[i]*deviceq[j]*zij)/(rij*rij*rij));

deviceFx[i] = deviceFx[i] + foxij;
deviceFy[i] = deviceFy[i] + foyij;
deviceFz[i] = deviceFz[i] + fozij;
deviceU[i] = deviceU[i] + 2*(deviceq[j]/rij);

}
}
}

}