#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_cdf( unsigned int* d_input_cdf, float*        d_output_cdf, int           n )
{
const float normalization_constant = 1.f / d_input_cdf[n - 1];

int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

if ( global_index_1d < n )
{
unsigned int input_value  = d_input_cdf[ global_index_1d ];
float        output_value = input_value * normalization_constant;

d_output_cdf[ global_index_1d ] = output_value;
}
}