#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initGuessBuffers( const uchar4* srcImg, float3* guess1, float3* guess2, const uint nRows, const uint nCols )
{
const uint nSamps = nRows*nCols;

const uint samp = threadIdx.x + blockDim.x * blockIdx.x;
if( samp < nSamps )
{
guess1[samp].x = srcImg[samp].x;
guess2[samp].x = srcImg[samp].x;

guess1[samp].y = srcImg[samp].y;
guess2[samp].y = srcImg[samp].y;

guess1[samp].z = srcImg[samp].z;
guess2[samp].z = srcImg[samp].z;
}
}