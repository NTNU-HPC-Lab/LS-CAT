#include "hip/hip_runtime.h"
#include "includes.h"
/*
* SpaceTime Simulator
*   Curso Deep Learning y Cuda - 2020
*   Autor: Oscar Noel Amaya Garcia
*   email: dbanshee@gmail.com
*/



#define RUN_MODE_SIM            0
#define RUN_MODE_BENCH          1

#define SP_FILENAME             "sp.json"
#define SP_FILENAME_BUFF1       "sp_0.json"
#define SP_FILENAME_BUFF2       "sp_1.json"
#define SP_FILENAME_BENCH       "sp_bench.json"
#define MAX_BLACK_HOLES         10
#define SOFTENING               1e-9f
#define DT                      0.05f

#define MAX_ASTEROIDS           30
#define AST_FILENAME            "ast.json"
#define AST_FILENAME_BUFF1      "ast_0.json"
#define AST_FILENAME_BUFF2      "ast_1.json"
#define AST_FILENAME_BENCH      "ast_bench.json"

#define MAX_BENCHMARKS          128
#define BENCH_FILENAME          "benchmark.json"
#define BENCH_TIME_SECS         10
#define BENCH_CPU               0
#define BENCH_GPU               1
#define BENCH_REGEN_BH_STEPS    5
#define BENCH_FILE_ACCESS_STEPS 3

#define CUDA_OPT_NLEVELS        4
#define CUDA_OPT_LEVEL_0        0
#define CUDA_OPT_LEVEL_1        1
#define CUDA_OPT_LEVEL_2        2
#define CUDA_OPT_LEVEL_3        3

#define MAX_TIME_SIMULATION_SEC 360
#define REGEN_BLACK_HOLES_SEC   20

#define NUM_BECHMARKS 10

typedef struct blackHole {
float x, y, g;
} blackHole;

typedef struct spacePoint {
float x, y, g;
} spacePoint;

typedef struct asteroid {
float x, y, vx, vy;
} asteroid;


typedef struct benchmark {
char name[1024];
int number;
int config;
int type;   // CPU = 0, GPU = 1
long time;  // millis
int steps;
} benchmark;


///////////////
// Global Vars
///////////////

// Runtime
int runMode = RUN_MODE_SIM;
int spCurrentBuff = 0;
int astCurrentBuff = 0;
int nBlackHoles = 0;
int nAsteroids = MAX_ASTEROIDS;
blackHole* blackHoles = NULL;
int bhSize;
asteroid* asteroids = NULL;
int astSize;
spacePoint* SPBox = NULL;
int spSize;
float top = 2, left = -2, bottom = -2, right = 2;
float spStep = 0.1;
int nelems;
int rows, cols;
int cudaOptLevel = CUDA_OPT_LEVEL_3;

// BenchMark
int nBenchmark;
int bechmarkRegenBHSteps = MAX_TIME_SIMULATION_SEC;
int bechmarkRegenWriteFileSteps = REGEN_BLACK_HOLES_SEC;
benchmark BENCHS[MAX_BENCHMARKS];
char benchName[1024];
int benchNum;
int benchConfig;
int benchType;


//////////////////
// Error Handling
//////////////////

__global__ void calculateAsteroidsKSHM(asteroid* AST, int nAsteroids, blackHole* BH, int nBlackHoles, float dt, int bhSize) {

extern __shared__ float s[];
blackHole* bhCache = (blackHole*) s;

int idx    = (blockIdx.x * blockDim.x) + threadIdx.x;
int stride = gridDim.x * blockDim.x;

if (idx == 0)
memcpy(bhCache, BH, bhSize);
__syncthreads();

for (int i = idx; i < nAsteroids; i += stride) {
float Fx = 0.0f; float Fy = 0.0f;

for (int j = 0; j < nBlackHoles; j++) {
float dx = AST[i].x - bhCache[j].x;
float dy = AST[i].y - bhCache[j].y;

if (dx == 0) break;
if (dy == 0) break;

float distSqr = dx*dx + dy*dy;
float invDist = 1 / sqrt(distSqr*distSqr);

Fx += dx * invDist; Fy += dy * invDist;
}

float modF = sqrt(Fx*Fx + Fy*Fy);
Fx = Fx / modF;
Fy = Fy / modF;

AST[i].vx += dt*-Fx;
AST[i].vy += dt*-Fy;
AST[i].x  += AST[i].vx*0.01;
AST[i].y  += AST[i].vy*0.01;
}
}