#include "hip/hip_runtime.h"
#include "includes.h"
/*
* SpaceTime Simulator
*   Curso Deep Learning y Cuda - 2020
*   Autor: Oscar Noel Amaya Garcia
*   email: dbanshee@gmail.com
*/



#define RUN_MODE_SIM            0
#define RUN_MODE_BENCH          1

#define SP_FILENAME             "sp.json"
#define SP_FILENAME_BUFF1       "sp_0.json"
#define SP_FILENAME_BUFF2       "sp_1.json"
#define SP_FILENAME_BENCH       "sp_bench.json"
#define MAX_BLACK_HOLES         10
#define SOFTENING               1e-9f
#define DT                      0.05f

#define MAX_ASTEROIDS           30
#define AST_FILENAME            "ast.json"
#define AST_FILENAME_BUFF1      "ast_0.json"
#define AST_FILENAME_BUFF2      "ast_1.json"
#define AST_FILENAME_BENCH      "ast_bench.json"

#define MAX_BENCHMARKS          128
#define BENCH_FILENAME          "benchmark.json"
#define BENCH_TIME_SECS         10
#define BENCH_CPU               0
#define BENCH_GPU               1
#define BENCH_REGEN_BH_STEPS    5
#define BENCH_FILE_ACCESS_STEPS 3

#define CUDA_OPT_NLEVELS        4
#define CUDA_OPT_LEVEL_0        0
#define CUDA_OPT_LEVEL_1        1
#define CUDA_OPT_LEVEL_2        2
#define CUDA_OPT_LEVEL_3        3

#define MAX_TIME_SIMULATION_SEC 360
#define REGEN_BLACK_HOLES_SEC   20

#define NUM_BECHMARKS 10

typedef struct blackHole {
float x, y, g;
} blackHole;

typedef struct spacePoint {
float x, y, g;
} spacePoint;

typedef struct asteroid {
float x, y, vx, vy;
} asteroid;


typedef struct benchmark {
char name[1024];
int number;
int config;
int type;   // CPU = 0, GPU = 1
long time;  // millis
int steps;
} benchmark;


///////////////
// Global Vars
///////////////

// Runtime
int runMode = RUN_MODE_SIM;
int spCurrentBuff = 0;
int astCurrentBuff = 0;
int nBlackHoles = 0;
int nAsteroids = MAX_ASTEROIDS;
blackHole* blackHoles = NULL;
int bhSize;
asteroid* asteroids = NULL;
int astSize;
spacePoint* SPBox = NULL;
int spSize;
float top = 2, left = -2, bottom = -2, right = 2;
float spStep = 0.1;
int nelems;
int rows, cols;
int cudaOptLevel = CUDA_OPT_LEVEL_3;

// BenchMark
int nBenchmark;
int bechmarkRegenBHSteps = MAX_TIME_SIMULATION_SEC;
int bechmarkRegenWriteFileSteps = REGEN_BLACK_HOLES_SEC;
benchmark BENCHS[MAX_BENCHMARKS];
char benchName[1024];
int benchNum;
int benchConfig;
int benchType;


//////////////////
// Error Handling
//////////////////

__global__ void calculateSpaceTimeKSHM(spacePoint* SPBox, int nRows, int nCols, float left, float right, float top, float bottom, blackHole* BH, int nBlackHoles, int bhSize) {

extern __shared__ float s[];
blackHole* bhCache = (blackHole*) s;

int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

if (i == 0 && j == 0)
memcpy(bhCache, BH, bhSize);
__syncthreads();

if (i < nRows && j < nCols) {
float x = (i / (float) nRows * (right-left)) + left;
float y = (j / (float) nCols * (top-bottom)) + bottom;

int idx = i*nRows+j;

float g = 0.0f;
for (int b = 0; b < nBlackHoles; b++){
float dx = x - bhCache[b].x;
float dy = y - bhCache[b].y;
float distSqr = sqrt(dx*dx + dy*dy);
if (distSqr == 0.0f) {
distSqr = 0.000000001;
}

float invDist = 1 / (pow((float)distSqr, (float)0.05));
g += (bhCache[b].g * invDist);
}

SPBox[idx].x = x;
SPBox[idx].y = y;
SPBox[idx].g = g;
}
}