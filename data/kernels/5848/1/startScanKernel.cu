#include "hip/hip_runtime.h"
#include "includes.h"
// risky
#define dfloat double

#define p_eps 1e-6

#define p_Nsamples 1

// ratio of importance in sampling primary ray versus random rays
#define p_primaryWeight 2.f

#define p_intersectDelta 0.1f

#define p_shadowDelta 0.15f
#define p_projectDelta 1e-2

#define p_maxLevel 5
#define p_maxNrays (2<<p_maxLevel)
#define p_apertureRadius 20.f
#define NRANDOM 10000

hipEvent_t startTimer, endTimer;

__global__ void startScanKernel(const int N, const int *v, int *scanv, int *starts){

__shared__ int s_v0[BLOCKSIZE];
__shared__ int s_v1[BLOCKSIZE];

int j = threadIdx.x;
int b = blockIdx.x;
int n = j + b*BLOCKSIZE;

s_v0[j] = (n<N) ?  v[j+b*BLOCKSIZE]: 0;

int offset = 1;
do{
__syncthreads();

s_v1[j] = (j<offset) ? s_v0[j] : (s_v0[j]+s_v0[j-offset]) ;

offset *= 2;

__syncthreads();

s_v0[j] = (j<offset) ? s_v1[j] : (s_v1[j]+s_v1[j-offset]) ;

offset *= 2;
} while(offset<BLOCKSIZE);

if(n<N)
scanv[n+1] = s_v0[j];

if(j==(BLOCKSIZE-1)){
starts[b+1] = s_v0[j];
}

}