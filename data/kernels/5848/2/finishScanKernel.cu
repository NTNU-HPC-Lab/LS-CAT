#include "hip/hip_runtime.h"
#include "includes.h"
// risky
#define dfloat double

#define p_eps 1e-6

#define p_Nsamples 1

// ratio of importance in sampling primary ray versus random rays
#define p_primaryWeight 2.f

#define p_intersectDelta 0.1f

#define p_shadowDelta 0.15f
#define p_projectDelta 1e-2

#define p_maxLevel 5
#define p_maxNrays (2<<p_maxLevel)
#define p_apertureRadius 20.f
#define NRANDOM 10000

hipEvent_t startTimer, endTimer;

__global__ void finishScanKernel(const int N, int *scanv, int *starts){

int j = threadIdx.x;
int b = blockIdx.x;

int n=j+b*BLOCKSIZE;

if(n<N){
int start = starts[b];

scanv[n+1] += start;
}
}