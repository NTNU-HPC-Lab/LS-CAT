#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binaryCrossEntropyCost(float* predictions, float* target, int size, float* cost) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
float partial_cost = target[index] * logf(predictions[index])
+ (1.0f - target[index]) * logf(1.0f - predictions[index]);
atomicAdd(cost, - partial_cost / size);
}
}