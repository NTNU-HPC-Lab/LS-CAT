#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x) {
return 1.0f / (1 + __expf(-x));
}
__global__ void sigmoidActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim) {
A[index] = sigmoid(Z[index]);
}
}