#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x) {
return 1.0f / (1 + __expf(-x));
}
__global__ void updateBiasKernel_sigmoid(float* dZ, float* b, int cols, int row, float learning_rate){
int bid = blockIdx.x;
extern __shared__ float _share[];
//float * _max = _share;
float * _sum = _share;
float* sp = dZ + cols * bid;
_sum[threadIdx.x] = 0.0;

for(int id = threadIdx.x ; id < cols; id += blockDim.x){
//	int id = tid + threadIdx.x;
//if(id < cols){
_sum[threadIdx.x] += sp[id];
//}
}
__syncthreads();
int len = blockDim.x;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
_sum[threadIdx.x] += _sum[threadIdx.x + skip];
}
len = (len + 1) >> 1;
}
__syncthreads();
b[bid] -= learning_rate * (_sum[0]/cols);
}