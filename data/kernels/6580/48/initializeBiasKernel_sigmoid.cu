#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x) {
return 1.0f / (1 + __expf(-x));
}
__global__ void initializeBiasKernel_sigmoid(float* b, int size){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < size){
b[index] = 0.0;
}
}