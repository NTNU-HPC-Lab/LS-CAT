#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_linear(float* softmaxP, float* b, int rows, int cols){
int tid = threadIdx.x;
int bid = blockIdx.x;

float _max = -100000000.0;
float sum = 0.0;

extern __shared__ float _share[];

if(tid * cols + bid < rows * cols){
for(int i = 0 ; i < rows ; i++) _share[i] = b[i * cols + bid];
for(int i = 0 ; i < rows ; i++)	_max = max(_max, _share[i]);
for(int i = 0 ; i < rows ; i++)	_share[i] = __expf(_share[i]-_max);
for(int i = 0 ; i < rows ; i++)	sum += _share[i];
for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] = _share[i]/sum;
}
}