#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyTanh(float* out, float* in1, float* in2, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size)
out[id] = in1[id] * in2[id];
}