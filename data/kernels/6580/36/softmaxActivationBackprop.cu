#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < Z_x_dim * Z_y_dim){
dZ[index] = dA[index];
}

}