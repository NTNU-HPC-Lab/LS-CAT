#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReluBackKernel(float* Z, float* dZ, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id < size){
if(Z[id] <= 0) dZ[id] = 0;
}
}