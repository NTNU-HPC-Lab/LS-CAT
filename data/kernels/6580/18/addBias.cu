#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addBias(float* Z, float* b, int Z_x_dim, int Z_y_dim){
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

if(row < Z_y_dim && col < Z_x_dim){
Z[row * Z_x_dim + col] += b[row];
}
}