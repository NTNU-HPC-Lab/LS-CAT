#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeBiasKernel_relu(float* b, int size){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < size){
b[index] = 0.0;
}
}