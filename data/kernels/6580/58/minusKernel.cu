#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minusKernel(float* A, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size){
A[id] = 1 - A[id];
}
}