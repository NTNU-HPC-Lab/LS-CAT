#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expKernel(float* Z, float* A, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size){
A[id] = __expf(-Z[id]);
}
}