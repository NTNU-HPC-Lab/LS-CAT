#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dMSECost(float* predictions, float* target, float* dY, int size) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
dY[index] = 2 * (predictions[index] - target[index]);
}
}