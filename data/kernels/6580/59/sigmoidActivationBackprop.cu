#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x) {
return 1.0f / (1 + __expf(-x));
}
__global__ void sigmoidActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim){
dZ[index] = dA[index] * sigmoid(Z[index]) * (1 - sigmoid(Z[index]));
}
}