#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divideTanh(float* out, float* in1, float* in2, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size)
out[id] = in1[id] / in2[id];
}