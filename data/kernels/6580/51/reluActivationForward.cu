#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim) {
A[index] = fmaxf(Z[index], 0);
}
}