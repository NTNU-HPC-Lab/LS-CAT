#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void oneMinusTanh(float* out, float* in, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size)
out[id] = 1 - in[id];
}