#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void linearLayerUpdateWeights(  float* dZ, float* A, float* W, int dZ_x_dim, int dZ_y_dim, int A_x_dim, int A_y_dim, float learning_rate) {

int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

// A is treated as transposed
int W_x_dim = A_y_dim;
int W_y_dim = dZ_y_dim;

float dW_value = 0.0f;

if (row < W_y_dim && col < W_x_dim) {
for (int i = 0; i < dZ_x_dim; i++) {
dW_value += dZ[row * dZ_x_dim + i] * A[col * A_x_dim + i];
}
W[row * W_x_dim + col] = W[row * W_x_dim + col] - learning_rate * (dW_value / A_x_dim);
}
}