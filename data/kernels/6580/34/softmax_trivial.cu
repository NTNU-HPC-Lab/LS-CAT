#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_trivial(float* softmaxP, float* b, int rows, int cols){
int tid = threadIdx.x;
int bid = blockIdx.x;

float _max = -100000000.0;
float sum = 0.0;

if(tid * cols + bid < rows * cols){
for(int i = 0 ; i < rows ; i++)	_max = max(_max, b[i * cols + bid]);
for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] = (b[i * cols + bid] - _max);
for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] = __expf(softmaxP[i * cols + bid]);
for(int i = 0 ; i < rows ; i++)	sum += softmaxP[i * cols + bid];
for(int i = 0 ; i < rows ; i++)	softmaxP[i * cols + bid] /= sum;
}
}