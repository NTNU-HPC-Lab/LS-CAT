#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void linearLayerBackprop(float* W, float* dZ, float *dA, int W_x_dim, int W_y_dim, int dZ_x_dim, int dZ_y_dim) {

int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

// W is treated as transposed
int dA_x_dim = dZ_x_dim;
int dA_y_dim = W_x_dim;

float dA_value = 0.0f;

if (row < dA_y_dim && col < dA_x_dim) {
for (int i = 0; i < W_y_dim; i++) {
dA_value += W[i * W_x_dim + row] * dZ[i * dZ_x_dim + col];
}
dA[row * dA_x_dim + col] = dA_value;
}
}