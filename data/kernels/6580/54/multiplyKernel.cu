#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyKernel(float* Z, float* A, float* B, int size){
int id = blockDim.x * blockIdx.x + threadIdx.x;

if(id < size){
Z[id] = A[id] * B[id];
}
}