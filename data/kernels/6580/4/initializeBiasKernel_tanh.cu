#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeBiasKernel_tanh(float* b, int size){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < size){
b[index] = 0.0;
}
}