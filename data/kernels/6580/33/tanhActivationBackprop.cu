#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanhActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim) {
float d = Z[index];
dZ[index] = dA[index] * (1 - d * d);
}
}