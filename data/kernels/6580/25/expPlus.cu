#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expPlus(float* out, float* in, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size)
out[id] = __expf(in[id]);
}