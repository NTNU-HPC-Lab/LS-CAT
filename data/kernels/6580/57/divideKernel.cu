#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divideKernel(float* A, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size){
A[id] = 1.0f / A[id];
}
}