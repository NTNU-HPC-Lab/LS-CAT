#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TanhBackKernel(float* Z, float* dZ, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id < size){
float t = (Z[id]);
dZ[id] = dZ[id] * (1-t*t) ;
}
}