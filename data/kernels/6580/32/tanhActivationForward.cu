#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanhActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim) {
A[index] = std::tanh(Z[index]);
}
}