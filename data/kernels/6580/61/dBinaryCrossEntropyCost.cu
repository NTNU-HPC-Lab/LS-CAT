#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int size) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
dY[index] = -1.0 * ( target[index]/predictions[index] - (1 - target[index])/(1 - predictions[index]) );
}
}