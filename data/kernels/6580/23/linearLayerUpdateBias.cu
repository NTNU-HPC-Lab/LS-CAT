#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void linearLayerUpdateBias(float* dZ, float* b, int dZ_x_dim, int dZ_y_dim, int b_x_dim, float learning_rate) {
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < dZ_x_dim * dZ_y_dim) {
int dZ_x = index % dZ_x_dim;
int dZ_y = index / dZ_x_dim;
atomicAdd(&b[dZ_y], - learning_rate * (dZ[dZ_y * dZ_x_dim + dZ_x] / dZ_x_dim));
}
}