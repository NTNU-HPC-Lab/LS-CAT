#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SigmoidBackKernel(float* Z, float* dZ, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id < size){
float t = Z[id];
dZ[id] = dZ[id] * t * (1-t) ;
}
}