#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void linearLayerForward( float* W, float* A, float* Z, float* b, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim) {

int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int Z_x_dim = A_x_dim;
int Z_y_dim = W_y_dim;

float Z_value = 0;

if (row < Z_y_dim && col < Z_x_dim) {
for (int i = 0; i < W_x_dim; i++) {
Z_value += W[row * W_x_dim + i] * A[i * A_x_dim + col];
}
Z[row * Z_x_dim + col] = Z_value + b[row];
}
}