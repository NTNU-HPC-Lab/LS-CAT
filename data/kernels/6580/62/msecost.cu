#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void msecost(float* predictions, float* target, int size, float* cost) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
float partial_cost = (predictions[index] - target[index]) * (predictions[index] - target[index]);
atomicAdd(cost,  partial_cost / size);
}
}