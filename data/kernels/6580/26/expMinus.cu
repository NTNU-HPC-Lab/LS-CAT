#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expMinus(float* out, float* in, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size)
out[id] = __expf(-in[id]);
}