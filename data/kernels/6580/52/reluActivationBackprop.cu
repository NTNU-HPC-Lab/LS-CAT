#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < Z_x_dim * Z_y_dim) {
if (Z[index] > 0) {
dZ[index] = dA[index];
}
else {
dZ[index] = 0;
}
}
}