#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x) {
return 1.0f / (1 + __expf(-x));
}
__global__ void transpose_sigmoid(float *odata, float *idata, int width, int height)
{
__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

// read the matrix tile into shared memory
// load one element per thread from device memory (idata) and store it
// in transpose_relud order in block[][]
unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
if((xIndex < width) && (yIndex < height))
{
unsigned int index_in = yIndex * width + xIndex;
block[threadIdx.y][threadIdx.x] = idata[index_in];
}

// synchronise to ensure all writes to block[][] have completed
__syncthreads();

// write the transpose_relud matrix tile to global memory (odata) in linear order
xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
if((xIndex < height) && (yIndex < width))
{
unsigned int index_out = yIndex * height + xIndex;
odata[index_out] = block[threadIdx.x][threadIdx.y];
}
}