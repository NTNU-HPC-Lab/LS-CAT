#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols, int row){
int bid = blockIdx.x;
extern __shared__ float _share[];
float * _max = _share;
float * _sum = _share + blockDim.x;
float* sp = softMaxP + bid;
_sum[threadIdx.x] = 0.0;
_max[threadIdx.x] = -100000000.0;
for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
//int id = tid + threadIdx.x;
//if(id < cols){
sp[tid] += b[tid];
_max[threadIdx.x] = max(_max[threadIdx.x], sp[tid]);
//}
}
__syncthreads();
int len = blockDim.x;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
if(_max[threadIdx.x] < _max[threadIdx.x + skip])
{
_max[threadIdx.x] = _max[threadIdx.x + skip];
}
}
len = (len + 1) >> 1;
}
__syncthreads();
for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
//	int id = tid + threadIdx.x;
//if(id < cols){
sp[tid] -= _max[0];
sp[tid] = __expf(sp[tid]);
_sum[threadIdx.x] += sp[tid];
//}
}
__syncthreads();
len = blockDim.x;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
_sum[threadIdx.x] += _sum[threadIdx.x + skip];
}
len = (len + 1) >> 1;
}
__syncthreads();
for(int tid = threadIdx.x * cols + blockIdx.x; tid < row * cols; tid += cols){
//int id = tid + threadIdx.x;
//if(id < cols){
sp[tid] /= _sum[0];
//}
}
}