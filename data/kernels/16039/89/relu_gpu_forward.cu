#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < N)
out[tid] = in[tid] > 0 ? in[tid] : 0;
}