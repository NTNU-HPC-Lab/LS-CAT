#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void golGpu(int height, int width, unsigned char* pBuffer1, unsigned char* pBuffer2){
int x = blockIdx.x * 2 + threadIdx.x;
int y = blockIdx.y * 2 + threadIdx.y;

int indx = x * height + y;

pBuffer2[indx] = pBuffer1[indx];

int num = 0;

if (x-1 >= 0 && x-1 < height && y >= 0 && y < width)
num += pBuffer1[(x-1) * height + y];

if (x+1 >= 0 && x+1 < height && y >= 0 && y < width)
num += pBuffer1[(x+1) * height + y];

if (x >= 0 && x < height && y-1 >= 0 && y-1 < width)
num += pBuffer1[x * height + (y-1)];

if (x >= 0 && x < height && y+1 >= 0 && y+1 < width)
num += pBuffer1[x * height + (y+1)];

if (x-1 >= 0 && x-1 < height && y-1 >= 0 && y-1 < width)
num += pBuffer1[(x-1) * height + (y-1)];

if (x-1 >= 0 && x-1 < height && y+1 >= 0 && y+1 < width)
num += pBuffer1[(x-1) * height + (y+1)];

if (x+1 >= 0 && x+1 < height && y-1 >= 0 && y-1 < width)
num += pBuffer1[(x+1) * height + (y-1)];

if (x+1 >= 0 && x+1 < height && y+1 >= 0 && y+1 < width)
num += pBuffer1[(x+1) * height + (y+1)];

if(num < 2)
pBuffer2[indx] = 0x0;

if(num > 3)
pBuffer2[indx] = 0x0;

if(num == 3 && !pBuffer1[indx])
pBuffer2[indx] = 0x1;
//return num;

}