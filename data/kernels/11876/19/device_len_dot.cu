#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_len_dot ()
{
__shared__ float partial_len[REDUC_THREADS], partial_dot[REDUC_THREADS] ;
int i, n, index ;
float sum_len, sum_dot ;

index = threadIdx.x ;
n = d_n_inputs_cols * d_nhid ;

sum_len = sum_dot = 0.0f ;
for (i=blockIdx.x*blockDim.x+index ; i<n ; i+=blockDim.x*gridDim.x) {
sum_len += d_w_grad[i] * d_w_grad[i] ;
sum_dot += d_w_grad[i] * d_prev_grad[i] ;
d_prev_grad[i] = d_w_grad[i] ;
}

partial_len[index] = sum_len ;
partial_dot[index] = sum_dot ;
__syncthreads() ;

for (i=blockDim.x>>1 ; i ; i>>=1) {
if (index < i) {
partial_len[index] += partial_len[index+i] ;
partial_dot[index] += partial_dot[index+i] ;
}
__syncthreads() ;
}

if (index == 0) {
d_len_out[blockIdx.x] = partial_len[0] ;
d_dot_out[blockIdx.x] = partial_dot[0] ;
}
}