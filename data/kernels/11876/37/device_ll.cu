#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_ll ()
{
__shared__ double partial_ll[REDUC_THREADS] ;
int i, n, ntarg, index ;
double sum_ll ;

index = threadIdx.x ;
n = d_ncases ;
ntarg = d_ntarg ;

sum_ll = 0.0 ;
for (i=blockIdx.x*blockDim.x+index ; i<n ; i+=blockDim.x*gridDim.x)
sum_ll -= log ( d_output[i*ntarg+d_class[i]] + 1.e-30 ) ;

partial_ll[index] = sum_ll ;
__syncthreads() ;

for (i=blockDim.x>>1 ; i ; i>>=1) {
if (index < i)
partial_ll[index] += partial_ll[index+i] ;
__syncthreads() ;
}

if (index == 0)
d_mse_out[blockIdx.x] = partial_ll[0] ;
}