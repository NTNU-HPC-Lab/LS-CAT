#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_transpose ()
{
int ivis, ihid ;

ivis = blockIdx.x * blockDim.x + threadIdx.x ;
if (ivis >= d_n_inputs)
return ;

ihid = blockIdx.y ;

d_wtr[ivis*d_nhid_cols+ihid] = d_w[ihid*d_n_inputs_cols+ivis] ;
}