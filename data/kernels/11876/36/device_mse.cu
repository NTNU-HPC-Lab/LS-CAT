#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_mse ()
{
__shared__ double partial_mse[REDUC_THREADS] ;
int i, index ;
unsigned int n ;
double diff, sum_mse ;

index = threadIdx.x ;
n = d_ncases * d_ntarg ;

sum_mse = 0.0 ;
for (i=blockIdx.x*blockDim.x+index ; i<n ; i+=blockDim.x*gridDim.x) {
diff = d_output[i] - d_targets[i] ;
sum_mse += diff * diff ;
}

partial_mse[index] = sum_mse ;
__syncthreads() ;

for (i=blockDim.x>>1 ; i ; i>>=1) {
if (index < i)
partial_mse[index] += partial_mse[index+i] ;
__syncthreads() ;
}

if (index == 0)
d_mse_out[blockIdx.x] = partial_mse[0] ;
}