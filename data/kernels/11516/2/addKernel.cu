#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int * dev_a, int * dev_b, int * dev_c)
{
int i = threadIdx.x;
dev_c[i] = dev_a[i] + dev_b[i];
}