#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int * dev_a, int* dev_b ,int* dev_size)
{
int i = threadIdx.x;
int j,p;
for (j = 0; j < (*dev_size); j++)
{
p = *dev_size*i + j;
dev_b[i] += dev_a[p];
//printf("%d %d\n", i, p);
}
}