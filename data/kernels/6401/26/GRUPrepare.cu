#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void finish(unsigned int* counter) {
__syncthreads();
__threadfence();
if (threadIdx.x == 0) { atomicAdd(counter, 1); }
}
__global__ void GRUPrepare(unsigned int* finished, const int round) {
for (int i = 0; i < round; i++) { finished[i] = 0; }
}