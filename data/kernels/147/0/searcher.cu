#include "hip/hip_runtime.h"
#include "includes.h"

#define COUNTERS 66
#define C_SIZE 64
#define C_STOP 65 // == C_SIZE+1
#define N 4224 // == COUNTERS*C_SIZE
#define N2 17842176 // == N*N

#define CUDA_ERROR_CHECK

#define cudaSafeCall(error) __cudaSafeCall(error, __FILE__, __LINE__)
#define cudaCheckErrors() __cudaCheckErrors(__FILE__, __LINE__)

__device__ inline int uniq(const int* M, int i, int* counters) {
for (int j = 1; j <= i - 1; j++) {
int a = (j - 1) * C_SIZE + counters[j-1];
int b = (i - 1) * C_SIZE + counters[i-1];
if (M[(a - 1) + N * (b - 1)] == 0)
return 1;
}
return 0;
}
__global__ void searcher(const int* M, int* res, size_t* itersNum) {
int partNumber = threadIdx.x + blockIdx.x * blockDim.x;
// initialize counters vector
int counters[COUNTERS];
for (int i = 0; i < COUNTERS; i++)
counters[i] = 1;

// go to selected part
counters[0] = 25;
counters[1] = 5;
counters[2] = 1;
counters[3] = 3;
counters[4] = 4;
counters[5] = 7;
counters[6] = 9;
counters[7] = 2;
counters[8] = 10;
counters[9] = 8;
counters[10] = (partNumber - 1) / 64 + 1;
counters[11] = (partNumber - 1) % 64 + 1;

size_t iter = 0;
size_t current = 1;
while (1) {
iter++;

// stop if search in the selected part is finished
if (counters[10] != (partNumber - 1) / 64 + 1 || counters[11] != (partNumber - 1) % 64 + 1) {
for (int i = 0; i < COUNTERS; i++)
res[partNumber * COUNTERS + i] = -1;
itersNum[partNumber] = iter;
break;
}

// first subspace is always good
if (current == 1)
current = 2;

// print intermediate state
// if (current == 13 && iter > 1000) {
//     fprintf(f, "Current state of part number %d:", partNumber);
//     for (int i = 0; i < COUNTERS; i++)
//         fprintf(f, " %d", counters[i]);
//     fprintf(f, "\nNumber of iterations: %f\n\n", iter);
//     fflush(f);
// }

for (int i = current; i <= COUNTERS; i++) {
if (uniq(M, i, counters) == 1) {
counters[i-1]++;
current = i;
while (counters[current-1] == C_STOP) {
counters[current - 1] = 1;
counters[current - 2] = counters[current - 2] + 1;
current--;
}
break;
}
}

if (current == COUNTERS && uniq(M, current, counters) == 0) {
for (int i = 0; i < COUNTERS; i++)
res[partNumber * COUNTERS + i] = counters[i];
itersNum[partNumber] = iter;
break;
}
}
}