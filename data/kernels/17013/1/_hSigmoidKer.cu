#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _hSigmoidKer(float const *in, float *out, int size) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index >= size)
return ;

if (in[index] > 3 )
out[index] = 1;
else if (in[index] < -3)
out[index] = 0;
else
out[index] = (in[index] + 3)/6;
}