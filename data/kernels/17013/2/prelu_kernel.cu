#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prelu_kernel(const float *input, float *output, int num_elem, int input_size, int fm_size, const float* gamma) {

int idx = threadIdx.x + blockDim.x * blockIdx.x;
if (idx >= num_elem) return;

if (input[idx] >= 0.0f) {
output[idx] = input[idx];
return;
}
int c = (idx % input_size) / fm_size;
output[idx] = input[idx] * gamma[c];
}