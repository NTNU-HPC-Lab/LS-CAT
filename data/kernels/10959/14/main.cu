#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "updateGradInputLSM.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *target = NULL;
hipMalloc(&target, XSIZE*YSIZE);
const float *mapping = NULL;
hipMalloc(&mapping, XSIZE*YSIZE);
const float *n_class_in_cluster = NULL;
hipMalloc(&n_class_in_cluster, XSIZE*YSIZE);
float *class_score = NULL;
hipMalloc(&class_score, XSIZE*YSIZE);
float *class_logsum = NULL;
hipMalloc(&class_logsum, XSIZE*YSIZE);
float *cluster_score = NULL;
hipMalloc(&cluster_score, XSIZE*YSIZE);
float *cluster_logsum = NULL;
hipMalloc(&cluster_logsum, XSIZE*YSIZE);
const long class_score_stride0 = 1;
const long cluster_score_stride0 = 1;
int n_clusters = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
updateGradInputLSM<<<gridBlock,threadBlock>>>(target,mapping,n_class_in_cluster,class_score,class_logsum,cluster_score,cluster_logsum,class_score_stride0,cluster_score_stride0,n_clusters);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
updateGradInputLSM<<<gridBlock,threadBlock>>>(target,mapping,n_class_in_cluster,class_score,class_logsum,cluster_score,cluster_logsum,class_score_stride0,cluster_score_stride0,n_clusters);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
updateGradInputLSM<<<gridBlock,threadBlock>>>(target,mapping,n_class_in_cluster,class_score,class_logsum,cluster_score,cluster_logsum,class_score_stride0,cluster_score_stride0,n_clusters);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}