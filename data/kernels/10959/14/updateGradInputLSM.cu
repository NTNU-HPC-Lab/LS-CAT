#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradInputLSM(const float* target, const float* mapping, const float* n_class_in_cluster, float* class_score, float* class_logsum, float* cluster_score, float* cluster_logsum, const long class_score_stride0, const long cluster_score_stride0, int n_clusters) {
const int tidx = threadIdx.x;
const int nthreads = blockDim.x;

const int itarget = (int)(target[blockIdx.x] - 0.5f);
const int cluster_target = (int)(mapping[2*itarget] - 0.5f);
const int idx_in_cluster_target = (int)(mapping[2*itarget+1] - 0.5f);
const int cluster_size = (int)(n_class_in_cluster[cluster_target] + 0.5f);

float *score, logsum_k, *target_score;
int N;
if (blockIdx.y == 0) {
score = cluster_score + blockIdx.x * cluster_score_stride0;
logsum_k = cluster_logsum[blockIdx.x];
N = n_clusters;
target_score = score + cluster_target;
} else {
score = class_score + blockIdx.x * class_score_stride0;
logsum_k = class_logsum[blockIdx.x];
N = cluster_size;
target_score = score + idx_in_cluster_target;
}

for (int i = tidx; i < N; i += nthreads)
score[i] = expf(score[i] - logsum_k);
__syncthreads(); //TODO : not exactly needed
if (tidx == 0)
*target_score -= 1.f;
}