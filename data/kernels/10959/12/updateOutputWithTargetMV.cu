#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateOutputWithTargetMV(const float* input, const float* weight, const float* bias, const float* mapping, const float* n_class_in_cluster, const float* class_start_indices, const float* target, const long input_stride0, const long weight_stride0, const long score_stride0, long input_size, float* score) {
__shared__ float buffer[MV_BUFFER_SIZE];
// align input and score to current sample in minibatch
input += input_stride0 * blockIdx.y;
score += score_stride0 * blockIdx.y;

// get the indices corresponding the the target
const int itarget = (int)(target[blockIdx.y] - 0.5f); // - 0.5 : 1based->0
const int cluster_target = (int)(mapping[2*itarget] - 0.5f);
const int iclass_start = (int)(class_start_indices[cluster_target] + 0.5f);
const int cluster_size = (int)(n_class_in_cluster[cluster_target] + 0.5f);

// get the bias and weight of the target cluster + correct line
const int lineIdx = blockIdx.x;
const int nLinesParallel = gridDim.x;

// do matrix vector multiply :
const int tidxx = threadIdx.x;
//   loop over lines
for (int iline = lineIdx; iline < cluster_size; iline += nLinesParallel) {
const float* weight0 = weight + weight_stride0 * (iclass_start + iline);
//   map
__syncthreads();
register float tmp = 0.f;
for (int i = tidxx; i < input_size; i += MV_BUFFER_SIZE)
tmp += input[i] * weight0[i];
buffer[tidxx] = tmp;
//   reduce
/*
for (unsigned int stride = MV_BUFFER_SIZE >> 1; stride > 0; stride >>= 1) {
__syncthreads();
if (tidxx < stride)
buffer[tidxx] += buffer[tidxx+stride];
}
if (tidxx == 0)
score[iline] = buffer[0] + bias[iclass_start + iline];
*/
tmp = 0.f;
__syncthreads();
if (tidxx < MV_BUFFER_SIZE / MV_N_REDUCE) {
for (int i = tidxx * MV_N_REDUCE; i < (tidxx + 1) * MV_N_REDUCE; ++i)
tmp += buffer[i];
buffer[tidxx] = tmp;
}
__syncthreads();
// store result
if (tidxx == 0) {
tmp = buffer[0];
#pragma unroll
for (int i = 1; i < MV_BUFFER_SIZE / MV_N_REDUCE; ++i)
tmp += buffer[i];
score[iline] = tmp + bias[iclass_start + iline];
}
}
}