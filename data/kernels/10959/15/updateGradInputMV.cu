#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradInputMV(const float* score, const float* weight, const float* mapping, const float* n_class_in_cluster, const float* class_start_indices, const float* target, const long gradInput_stride0, const long weight_stride0, const long score_stride0, int input_size, float* gradInput) {
// align input and score to current sample in minibatch
gradInput += gradInput_stride0 * blockIdx.y;
score += score_stride0 * blockIdx.y;

// get the indices corresponding the the target
const int itarget = (int)(target[blockIdx.y] - 0.5f); // - 0.5 : 1based->0
const int cluster_target = (int)(mapping[2*itarget] - 0.5f);
const int iclass_start = (int)(class_start_indices[cluster_target] + 0.5f);
const int cluster_size = (int)(n_class_in_cluster[cluster_target] + 0.5f);

// get the bias and weight of the target cluster + correct line
const int colIdx = blockIdx.x * MV2_NLINES + threadIdx.x;
const int nColParallel = gridDim.x * MV2_NLINES;

//   loop over lines
weight += weight_stride0 * iclass_start;
for (int icol = colIdx; icol < input_size; icol += nColParallel) {
const float* weight0 = weight + icol;
//   map
register float tmp = 0.f;
for (int i = 0; i < cluster_size; ++i)
tmp += score[i] * weight0[weight_stride0 * i];
gradInput[icol] = tmp;
}
}