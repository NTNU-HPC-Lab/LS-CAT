#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TemporalConvolutionTBC_fp_bias( float* output_features, float* bias, int output_stride, int rows) {
int x = blockIdx.x * 32 + threadIdx.x;
float b = bias[x];
for (int row = blockIdx.y; row < rows; row += gridDim.y) {
output_features[row * output_stride + x] = b;
}
}