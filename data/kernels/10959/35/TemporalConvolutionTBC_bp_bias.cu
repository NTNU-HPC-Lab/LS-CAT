#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TemporalConvolutionTBC_bp_bias( float* matrix, float* target, int rows, int stride, float scale) {
int i = blockIdx.x * 32 + threadIdx.x;
float t = 0;
for (int j = blockIdx.y; j < rows; j += gridDim.y)
t += matrix[j * stride + i];
atomicAdd(&target[i], t * scale);
}