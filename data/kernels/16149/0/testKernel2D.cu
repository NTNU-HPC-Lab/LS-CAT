#include "hip/hip_runtime.h"
#include "includes.h"
/*
Autor: Munesh Singh
Date: 08 March 2010
Vector addition using hipMallocPitch
*/

const int width = 567;
const int height = 985;


__global__ void testKernel2D(float* M, float* N, float* P, size_t pitch) {
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;
if (row < width && col < width) {
float* row_M = (float*)((char*)M + row * pitch);
float* row_N = (float*)((char*)N + row * pitch);
float* row_P = (float*)((char*)P + row * pitch);

row_P[col] = row_M[col] + row_N[col];
}
}