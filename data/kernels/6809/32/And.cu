#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void And( bool * x, size_t idx, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
x[(idx-2)*N + i] = x[(idx-2)*N + i] & x[(idx-1)*N + i];
}
return;
}