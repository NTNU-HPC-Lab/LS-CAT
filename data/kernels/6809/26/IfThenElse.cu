#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IfThenElse(bool * b, float * x, size_t idxb, size_t idxf, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
//printf("From IfThenElse %d, %f, %f\n", b[(idxb-1)*N+i], x[(idxf-1)*N+i], x[(idxf-2)*N+i]);
if (b[(idxb-1)*N+i])
x[(idxf-2)*N+i] = x[(idxf-1)*N+i];
//printf("After IfThenElse %f\n", x[(idxf-2)*N+i]);

}
return;
}