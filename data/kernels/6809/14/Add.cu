#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float * x, size_t idx, size_t N, float W0, float W1)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
//printf("Adding %f and %f\n",x[(idx-1)*N + i], x[(idx-2)*N + i]);
//printf("idx = %d, N = %d, i = %d\n", idx, N, i);
//printf("%f %f %f %f %f %f\n", x[0], x[1], x[2], x[3], x[4], x[5]);
x[(idx-2)*N + i] = x[(idx-1)*N + i]*W0 + x[(idx-2)*N + i]*W1;
//printf("on stack %f\n",x[(idx-2)*N + i]);
//printf("%f %f %f %f\n", x[0], x[1], x[2], x[3]);//, x[4], x[5]);
}
return;
}