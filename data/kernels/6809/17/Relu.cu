#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Relu(float * x, size_t idx, size_t N, float W0)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
x[(idx-1)*N + i] = W0*x[(idx-1)*N + i] > 0 ? W0*x[(idx-1)*N + i] : 0.01;

return;
}