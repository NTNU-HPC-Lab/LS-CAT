#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void If(bool * xb, float * xf, size_t idxf, size_t idxb, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
/* if (xb[idx-1]) */
/*     xf[idx-1] = xf[idx-1]; */
/* else */
/*     out[i] = 0; */
if (!xb[(idxb-1)*N+i])
xf[(idxf-1)*N+i] = 0;
}
return;
}