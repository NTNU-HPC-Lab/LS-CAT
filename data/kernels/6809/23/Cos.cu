#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Cos( float * x, size_t idx, size_t N, float W0)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
x[(idx-1)*N+i] = cos ( W0*x[(idx-1)*N+i] );
}
return;
}