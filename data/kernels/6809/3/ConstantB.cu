#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConstantB(bool * x, bool value, size_t idx, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
x[(idx)*N + i] = value;
}
return;
}