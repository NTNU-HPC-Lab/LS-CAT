#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LessThan(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
//printf("From less than %f %f %d \n", xf[(idxf-1)*N+i], xf[(idxf-2)*N+i], xf[(idxf-1)*N+i] < xf[(idxf-2)*N+i]);
xb[idxb*N+i] = (xf[(idxf-1)*N+i] < xf[(idxf-2)*N+i]);
}
return;
}