#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Sign( float * x, size_t idx, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
float res = x[(idx-1)*N+i];
if (res > 0 )
x[(idx-1)*N+i] = 1.0 ;
else if (res == 0)
x[(idx-1)*N+i] = 0.0;
else
x[(idx-1)*N+i] = -1.0 ;
}
return;
}