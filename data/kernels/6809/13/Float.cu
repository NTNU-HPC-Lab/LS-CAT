#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Float(float * x, int* y, size_t idxf, size_t idxi, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
x[(idxf)*N + i] = float(y[(idxi-1)*N + i]);

return;
}