#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Float(float * x, bool* y, size_t idxf, size_t idxb, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
x[(idxf)*N + i] = float(y[(idxb-1)*N + i]);

return;
}