#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Not( bool * x, size_t idx, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
x[(idx-1)*N+i] = ! x[(idx-1)*N+i] ;
}
return;
}