#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract( float * x, size_t idx, size_t N, float W0, float W1)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
x[(idx-2)*N+i] = W0*x[(idx-1)*N+i] - W1*x[(idx-2)*N+i];
}
return;
}