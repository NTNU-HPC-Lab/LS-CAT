#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Split(int * xi, bool * xb, size_t idxi, size_t idxb, size_t N, float threshold)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
xb[(idxb)*N+i] = (((float)xi[(idxi-1)*N+i]) == threshold);
}
return;
}