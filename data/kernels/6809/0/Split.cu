#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Split(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N, float threshold)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
xb[(idxb)*N+i] = (xf[(idxf-1)*N+i] < threshold);
}
return;
}