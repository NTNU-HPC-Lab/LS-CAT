#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LEQ(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
{
xb[idxb*N+i] = xf[(idxf-1)*N+i] <= xf[(idxf-2)*N+i];
}
return;
}