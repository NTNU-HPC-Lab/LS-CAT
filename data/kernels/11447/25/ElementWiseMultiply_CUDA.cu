#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ElementWiseMultiply_CUDA(double *C, double *A, double *B, int rows, int cols)
{
int j = blockDim.x * blockIdx.x + threadIdx.x;
int i = blockDim.y * blockIdx.y + threadIdx.y;

int sourceLength = cols * rows;
int sourceIndex = i + (j * blockDim.y);
int targetIndex = i + (j * blockDim.y);
if ((sourceIndex <= sourceLength - 1) & (targetIndex < rows))
{
//if (i == 0 & j == 0)
//{
//	printf("ElementWiseMultiply_CUDA, matrix A:\r\n");
//	printMatrix_CUDA << <1, 1 >> > (A, dimA);
//	printf("ElementWiseMultiply_CUDA, matrix B:\r\n");
//	printMatrix_CUDA << <1, 1 >> > (B, dimB);
//}
//int idx = i + (j * dimC.y);
double a = A[sourceIndex];
double b = B[sourceIndex];
C[targetIndex] = a * b;
//printf("i=%i, j=%i idx=%i | %i = %i * %i\r\n", i, j, idx, C[idx], a, b);
}
}