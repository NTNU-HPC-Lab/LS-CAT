#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
unsigned int i, ixj; /* Sorting partners: i and ixj */
i = threadIdx.x + blockDim.x * blockIdx.x;
ixj = i^j;

/* The threads with the lowest ids sort the array. */
if ((ixj)>i) {
if ((i&k)==0) {
/* Sort ascending */
if (dev_values[i]>dev_values[ixj]) {
/* exchange(i,ixj); */
float temp = dev_values[i];
dev_values[i] = dev_values[ixj];
dev_values[ixj] = temp;
}
}
if ((i&k)!=0) {
/* Sort descending */
if (dev_values[i]<dev_values[ixj]) {
/* exchange(i,ixj); */
float temp = dev_values[i];
dev_values[i] = dev_values[ixj];
dev_values[ixj] = temp;
}
}
}
}