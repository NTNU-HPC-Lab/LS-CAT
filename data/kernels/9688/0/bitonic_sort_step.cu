#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS 256
#define BLOCKS 32
#define NUM THREADS*BLOCKS

int seed_var =1239;

__device__ void swap(int *xp, int *yp)
{
int temp = *xp;
*xp = *yp;
*yp = temp;
}
__global__ void bitonic_sort_step(int *d_pr, int *d_bt, int j, int k)
{
int i, ixj; /* Sorting partners: i and ixj */
i = threadIdx.x + blockDim.x * blockIdx.x;
ixj = i^j;

/* The threads with the lowest ids sort the array. */
if ((ixj)>i)
{
if ((i&k)==0)
{
/* Sort ascending */
if (d_pr[i]>d_pr[ixj])
{
/* exchange(i,ixj); */
swap(&d_pr[i],&d_pr[ixj]);
swap(&d_bt[i],&d_bt[ixj]);
}
}
if ((i&k)!=0)
{
/* Sort descending */
if (d_pr[i]<d_pr[ixj])
{
/* exchange(i,ixj); */
swap(&d_pr[i], &d_pr[ixj]);
swap(&d_bt[i], &d_bt[ixj]);
}
}
}
}