#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS 256
#define BLOCKS 32
#define NUM THREADS*BLOCKS

int seed_var =1239;

__global__ void work_efficient_scan_kernel(int *X, int *Y, int InputSize)
{
extern __shared__ int XY[];
int i= blockIdx.x*blockDim.x+ threadIdx.x;
if (i < InputSize)
{
XY[threadIdx.x] = X[i];
}
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
{
__syncthreads();
int index = (threadIdx.x+1) * 2* stride -1;
if (index < blockDim.x)
{
XY[index] += XY[index -stride];
}
}
for (int stride = THREADS/4; stride > 0; stride /= 2)
{
__syncthreads();
int index = (threadIdx.x+1)*stride*2 -1;
if(index + stride < THREADS)
{
XY[index + stride] += XY[index];
}
}
__syncthreads();
Y[i] = XY[threadIdx.x];

//OWN CODE
__syncthreads();
if(threadIdx.x < blockIdx.x)
{
XY[threadIdx.x] = Y[threadIdx.x*blockDim.x + (blockDim.x-1)];
}
__syncthreads();
for(unsigned int stride =0; stride < blockIdx.x; stride++)
{
Y[i] += XY[stride];
}
__syncthreads();
}