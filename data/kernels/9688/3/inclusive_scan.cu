#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inclusive_scan(const unsigned int *X, unsigned int *Y, int N)
{
extern __shared__ int XY[];
unsigned   int i = blockIdx.x * blockDim.x + threadIdx.x;
// load input into __shared__ memory
if(i<N)
{
XY[threadIdx.x] =X[i];
}
/*Note here stride <= threadIdx.x, means that everytime the threads with threadIdx.x less than
stride do not participate in loop*/
for(unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
__syncthreads();
XY[threadIdx.x]+= XY[threadIdx.x - stride];
}
/*This is executed by all threads, so that they store the final prefix sum to
corresponding locations in global   memory*/
Y[i]=XY[threadIdx.x];

// wait until all threads of this block writes the output for all prefix sum within the block
__syncthreads();
if (threadIdx.x < blockIdx.x) //for 1st block onwards
{
//update the shared memory to keep prefix sum of last elements of previous block's
XY[threadIdx.x] = Y[threadIdx.x * blockDim.x + BLOCK_SIZE - 1];
}
__syncthreads();
for (int stride = 0; stride < blockIdx.x; stride++)
{    //add all previous las elements to this block elements
Y[threadIdx.x + blockDim.x * blockIdx.x] += XY[stride];
__syncthreads();

}
}