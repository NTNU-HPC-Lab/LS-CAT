#include "hip/hip_runtime.h"
#include "includes.h"
/*
sergeim19
April 27, 2015
Burgers equation - GPU CUDA version
*/


#define NADVANCE (4000)
#define nu (5.0e-2)

__global__ void kernel_calc_uu(double *u_dev, double *uu_dev)
{
int j;
j = blockIdx.x * blockDim.x + threadIdx.x;

uu_dev[j] = 0.5 * u_dev[j] * u_dev[j];
}