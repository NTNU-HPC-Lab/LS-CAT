#include "hip/hip_runtime.h"
#include "includes.h"
/*
sergeim19
April 27, 2015
Burgers equation - GPU CUDA version
*/


#define NADVANCE (4000)
#define nu (5.0e-2)

__global__ void kernel_rescale_u(double *u_dev, int N)
{
int j;
j = blockIdx.x * blockDim.x + threadIdx.x;

u_dev[j] = u_dev[j] / (double)N;
}