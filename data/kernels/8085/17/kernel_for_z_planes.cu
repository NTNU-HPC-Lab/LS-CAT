#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void Device_FloodFillZPlane(int zPlane, int L, int M, int N, unsigned char* vol)
{
long idx, idxS, idxN, ts;
bool anyChange = false;
int x, y;

ts = L*M*N;
// set point (0,0) to OUTSIZE_1
idx = zPlane*L*M /* + 0*L + 0 */;
vol[idx] = OUTSIDE_1;

anyChange = true;
while(anyChange) {

anyChange = false;
// loop from left to right and top to bottom
for(x=0; x < L; x++) {
for(y=0; y < M; y++) {
idxS = idx + y*L + x;
// if the point is set to OUTSIDE_1, the set all empty neightbors
// to OUTSIDE_1
if(vol[idxS] == OUTSIDE_1) {

idxN = idxS + L;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS - L;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS + 1;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS - 1;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}
}
}
}

if(anyChange) {
// same loop but bottom to top and right to left
anyChange = false;
// loop from left to right and top to bottom
for(x=L-1; x >=0; x--) {
for(y=M-1; y >=0; y--) {
idxS = idx + y*L + x;
// if the point is set to OUTSIDE_1, the set all empty neightbors
// to OUTSIDE_1
if(vol[idxS] == OUTSIDE_1) {

idxN = idxS + L;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS - L;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS + 1;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}

idxN = idxS - 1;
if((idxN >= 0) && (idxN < ts) && (vol[idxN] == 0)) {
vol[idxN] = OUTSIDE_1;
anyChange = true;
}
}
}
}
}
}


}
__global__ void kernel_for_z_planes(unsigned char *d_vol,int L,int M,int N)
{
Device_FloodFillZPlane(threadIdx.x,L,M,N,d_vol);
}