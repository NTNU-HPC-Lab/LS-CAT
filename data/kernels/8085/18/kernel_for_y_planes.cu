#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void Device_FloodFillYPlane(int yPlane, int L, int M, int N, unsigned char* vol)
{
long idx, idxS, idxN, ts;
bool anyChange = false;
int x, z;

ts = L*M*N;
// set point (0,0) to OUTSIZE_2
idx = /*0*L*M  + */ yPlane*L /*+ 0 */;
vol[idx] = OUTSIDE_2;

anyChange = true;
while(anyChange) {

anyChange = false;
// loop from left to right and top to bottom
for(x=0; x < L; x++) {
for(z=0; z < N; z++) {
idxS = z*L*M + idx + x;
// if the point is set to OUTSIDE_2, the set all empty neightbors
// to OUTSIDE_2
if(vol[idxS] == OUTSIDE_2) {

idxN = idxS + L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS - L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS + 1;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS - 1;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}
}
}
}

if(anyChange) {
// same loop but bottom to top and right to left

anyChange = false;
// loop from left to right and top to bottom
for(x=L-1; x >= 0; x--) {
for(z=N-1; z >= 0; z--) {
idxS = z*L*M + idx + x;
// if the point is set to OUTSIDE_2, the set all empty neightbors
// to OUTSIDE_2
if(vol[idxS] == OUTSIDE_2) {

idxN = idxS + L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS - L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS + 1;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}

idxN = idxS - 1;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1))) {
vol[idxN] = OUTSIDE_2;
anyChange = true;
}
}
}
}
}
}



}
__global__ void kernel_for_y_planes(unsigned char *d_vol,int L,int M,int N)
{
Device_FloodFillYPlane(threadIdx.x,L,M,N,d_vol);
}