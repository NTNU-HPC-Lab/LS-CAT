#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void Device_FloodFillXPlane(int xPlane, int L, int M, int N, unsigned char* vol)
{

long idx, idxS, idxN, ts;
bool anyChange = false;
int y, z;

ts = L*M*N;
// set point (0,0) to OUTSIZE_3
idx = /*0*L*M  +  yPlane*L */+ xPlane ;
vol[idx] = OUTSIDE_3;

anyChange = true;
while(anyChange) {

anyChange = false;
// loop from left to right and top to bottom
for(y=0; y < M; y++) {
for(z=0; z < N; z++) {
idxS = z*L*M + L*y + idx;
// if the point is set to OUTSIDE_3, the set all empty neightbors
// to OUTSIDE_3
if(vol[idxS] == OUTSIDE_3) {

idxN = idxS + L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS - L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS + L;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS - L;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}
}
}
}

if(anyChange) {
// same loop but bottom to top and right to left

anyChange = false;
// loop from left to right and top to bottom
for(y=M-1; y >= 0; y--) {
for(z=N-1; z >= 0; z--) {
idxS = z*L*M + + L*y + idx;
// if the point is set to OUTSIDE_3, the set all empty neightbors
// to OUTSIDE_3
if(vol[idxS] == OUTSIDE_3) {

idxN = idxS + L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1)  || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS - L*M;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS + L;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}

idxN = idxS - L;
if((idxN >= 0) && (idxN < ts) &&
((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) {
vol[idxN] = OUTSIDE_3;
anyChange = true;
}
}
}
}
}
}



}
__global__ void kernel_for_x_planes(unsigned char *d_vol,int L,int M,int N)
{
Device_FloodFillXPlane(threadIdx.x,L,M,N,d_vol);
}