#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
// you are guaranteed this is called with <<<1, 64, 64*4>>>
extern __shared__ float sdata[];
int tid  = threadIdx.x;
sdata[tid] = d_in[tid];
__syncthreads();

for (int stage = 0; stage <= 5; stage++)
{
for (int substage = stage; substage >= 0; substage--)
{
// TODO
}
}

d_out[tid] = sdata[tid];
}