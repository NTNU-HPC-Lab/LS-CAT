#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resized(unsigned char *imgData, int width, float scale_factor, hipTextureObject_t texObj) {
const unsigned  int tidX = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned  int tidY = blockIdx.y * blockDim.y + threadIdx.y;

const unsigned idx = tidY * width + tidX;

//Read texture mem to CUDA Kernel

imgData[idx] = tex2D<unsigned char>(texObj,(float)(tidX*scale_factor),(float)(tidY*scale_factor));

}