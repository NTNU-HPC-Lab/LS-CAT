#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int n, float *x, float *y) {

int lane_id = threadIdx.x & 31;
size_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
size_t warps_per_grid = (blockDim.x * gridDim.x) >> 5;
size_t warp_total = ((sizeof(float)*n) + STRIDE_64K-1) / STRIDE_64K;


if(blockIdx.x==0 && threadIdx.x==0) {
//printf("\n TId[%d] ", threadIdx.x);
//printf(" WId[%u] ", warp_id);
//printf(" LId[%u] ", lane_id);
//printf(" WperG[%u] ", warps_per_grid);
//printf(" wTot[%u] ", warp_total);
//printf(" rep[%d] ", STRIDE_64K/sizeof(float)/32);
}
for(; warp_id < warp_total; warp_id += warps_per_grid) {
#pragma unroll
for(int rep = 0; rep < STRIDE_64K/sizeof(float)/32; rep++) {
size_t ind = warp_id * STRIDE_64K/sizeof(float) + rep * 32 + lane_id;
if (ind < n) {
x[ind] = 1.0f;
//if(blockIdx.x==0 && threadIdx.x==0) {
//	printf(" \nind[%d] ", ind);
//}
y[ind] = 2.0f;
}
}
}

}