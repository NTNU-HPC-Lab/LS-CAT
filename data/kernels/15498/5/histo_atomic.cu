#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_atomic(unsigned int *out_histo,const float *d_in, int num_bins, int size,float min_val,float range)
{
int tid = threadIdx.x;
int id = tid + blockIdx.x * blockIdx.x;
if(tid >= size)
{
return;
}
int bin = ((d_in[id]-min_val)*num_bins)/range;
bin = bin == num_bins ? num_bins -1 : bin; //max value bin is last bin of the histogram
atomicAdd(&(out_histo[bin]),1);
}