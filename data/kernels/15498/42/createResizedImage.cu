#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj)
{
const unsigned int tidX = blockIdx.x*blockDim.x + threadIdx.x;
const unsigned int tidY = blockIdx.y*blockDim.y + threadIdx.y;
const unsigned index = tidY*scaled_width+tidX;

//Step 3: Read the texture memory from your texture reference in CUDA Kernel
imageScaledData[index] = tex2D<unsigned char>(texObj,(float)(tidX*scale_factor),(float)(tidY*scale_factor));
}