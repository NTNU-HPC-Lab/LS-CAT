#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int n, float *x, float *y) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride) {
x[i] = 1.0f;
y[i] = 2.0f;
}
}