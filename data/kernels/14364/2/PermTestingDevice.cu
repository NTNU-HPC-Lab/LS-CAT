#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PermTestingDevice(int numPermutations, int N, int V, int N_gp1, double *dataDevice, int *permutationsDevice, double *MaxTDevice)
{
int threadId = threadIdx.x + (blockIdx.x * blockDim.x); // Current Permutation
printf("Starting thread: %d \n", threadId);
int permutationsStart = threadId * N;
int i,j;
int N_gp2 = N - N_gp1; // Size of group 2
double group1Sum = 0;
double group2Sum = 0;
double group1SumSquared = 0; // Sum of all terms squared of group1 : x1^2 + x2^2 ...
double group2SumSquared = 0; // Sum of all terms squared of group2
double group1Mean = 0;
double group2Mean = 0;
double group1Var = 0;
double group2Var = 0;
double meanDifference = 0; // t-statistics numerator
double denominator = 0; // t-statistic denominator
double MaxT = 0;
double tStat = 0;

double voxelStatistic = 0;
int currSubject = 0;

/* For each voxel calculate a t-statistic*/
for(i = 0; i < V;i++)
{
group1Sum = 0;
group2Sum = 0;
group1SumSquared = 0;
group2SumSquared = 0;

/* Add statistics of the first group */
for(j = 0;j < N_gp1;j++)
{
currSubject = permutationsDevice[permutationsStart + j] - 1;
voxelStatistic = dataDevice[currSubject*V];
group1Sum = group1Sum + voxelStatistic;
group1SumSquared = group1SumSquared + voxelStatistic*voxelStatistic;
}

/* Add statistics of second group */
for(j = N_gp1; j < N; j++)
{
currSubject = permutationsDevice[permutationsStart + j] - 1;
voxelStatistic = dataDevice[currSubject*V];
group2Sum = group2Sum + voxelStatistic;
group2SumSquared = group2SumSquared + voxelStatistic*voxelStatistic;
}

group1Mean = group1Sum/N_gp1;
group2Mean = group2Sum/N_gp2;

group1Var = (group1SumSquared/N_gp1) - (group1Mean*group1Mean);
group2Var = (group2SumSquared/N_gp2) - (group2Mean*group2Mean);

meanDifference = group1Mean - group2Mean;
denominator = sqrt((group1Var / N_gp1) + (group2Var / N_gp2));

tStat = meanDifference/denominator;
if(tStat > MaxT)
{
MaxT = tStat;
}
}

MaxTDevice[threadId] = MaxT;
}