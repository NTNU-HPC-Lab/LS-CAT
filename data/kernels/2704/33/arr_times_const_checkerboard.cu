#include "hip/hip_runtime.h"
#include "includes.h"
/************************* CudaMat ******************************************
*   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
*   heintzmann@gmail.com                                                  *
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; Version 2 of the License.               *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program; if not, write to the                         *
*   Free Software Foundation, Inc.,                                       *
*   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
***************************************************************************
* Compile with:
* Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
* File sudo vi /usr/local/cuda/bin/nvcc.profile
* needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
*/

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177


#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


__global__ void arr_times_const_checkerboard(float*a,float b, float * c, size_t N, size_t sx,size_t sy,size_t sz)
{
size_t ids=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); // which source array element do I have to deal with?
if(ids>=N) return;  // not in range ... quit

size_t px=(ids/2)%sx;   // my x pos
size_t py=(ids/2)/sx;   // my y pos
float minus1=(1-2*((px+py)%2));
c[ids]=a[ids]*b*minus1;
}