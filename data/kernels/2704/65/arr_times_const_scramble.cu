#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arr_times_const_scramble(float*a,float b, float * c, int sx,int sy,int sz, int ox, int oy, int oz)
{
int pnum=blockIdx.x*blockDim.x+threadIdx.x;   // which source array element do I have to deal with?

int px=pnum%(sx/2);   // my x pos of a complex number in the subarray
int py=pnum/(sx/2);   // my y pos of a complex number
if(px>=(sx/2) || py >= (sy/2)) return;  // not in range ... quit
int ids=2*(px+py*sx);  /// offset to array start in floats
int idd=2*((ox+px)+(oy+py)*sx);

// echange two values using a tmp
float tmpR = c[idd];
float tmpI = c[idd+1];
c[idd]=a[ids]; // (float)(ox+px); //
c[idd+1]=a[ids+1]; // (float)(oy+py); //
a[ids]=tmpR;
a[ids+1]=tmpI;
}