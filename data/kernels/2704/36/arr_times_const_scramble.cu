#include "hip/hip_runtime.h"
#include "includes.h"
/************************* CudaMat ******************************************
*   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
*   heintzmann@gmail.com                                                  *
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; Version 2 of the License.               *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program; if not, write to the                         *
*   Free Software Foundation, Inc.,                                       *
*   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
***************************************************************************
* Compile with:
* Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
* File sudo vi /usr/local/cuda/bin/nvcc.profile
* needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
*/

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177


#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


__global__ void arr_times_const_scramble(float*a,float b, float * c, size_t sx,size_t sy,size_t sz, size_t ox, size_t oy, size_t oz)
{
// which source array element do I have to deal with?
size_t pnum=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);

size_t px=pnum%(sx/2);   // my x pos of a complex number in the subarray
size_t py=pnum/(sx/2);   // my y pos of a complex number
if(px>=(sx/2) || py >= (sy/2)) return;  // not in range ... quit
size_t ids=2*(px+py*sx);  /// offset to array start in floats
size_t idd=2*((ox+px)+(oy+py)*sx);

// echange two values using a tmp
float tmpR = c[idd];
float tmpI = c[idd+1];
c[idd]=a[ids]; // (float)(ox+px); //
c[idd+1]=a[ids+1]; // (float)(oy+py); //
a[ids]=tmpR;
a[ids+1]=tmpI;
}