#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate(float*a,float b, float * c, int sx,int sy,int sz, int dx, int dy, int dz, int ux, int uy, int uz)
{
int id=(blockIdx.x*blockDim.x+threadIdx.x); // id of this processor

int Processes=blockDim.x * gridDim.x;
int chains=ux*uy*uz; // total number of independent chains
int N=sx*sy*sz;  // total size of array, has to be chains*length_of_chain
int length=N/chains;  // chain length
int steps=N/Processes;  // this is how many steps each processor has to do

int step,nl,nx,ny,nz,x,y,z,i,idd;
float swp, nswp;

//if (id != 0)   return;
//for (id=0;id<Processes;id++)
{
step=steps*id;   // my starting step as the id times the number of steps
nl=step%length;  // current position in chain length
nx=(step/length)%ux;  // current position in unit cell x
ny=(step/(length*ux))%uy;  // current position in unit cell y
nz=(step/(length*ux*uy))%uz;  // current position in unit cell z
i=0;

//if (step/steps != 4 && step/steps != 5) return;

while(nz<uz)
{
while(ny<uy)
{
while (nx<ux)
{
x=(nx+nl*dx)%sx;  // advance by the offset steps along the chain
y=(ny+nl*dy)%sy;
z=(nz+nl*dz)%sz;
idd=x+sx*y+sx*sy*z;
if (i < steps) {
swp=a[idd];
// a[idd]=a[idd]+0.1;
__syncthreads();
}
while (nl<length-1)
{
if (i > steps-1)
goto nextProcessor; // return;
if (step >= N)  // this thread has reached the end of the total data to process
goto nextProcessor; // return;
step++;
x = (x+dx)%sx; // new position
y = (y+dy)%sy;
z = (z+dz)%sz;
idd=x+sx*y+sx*sy*z;
if (i < steps-1) {
nswp=a[idd];
__syncthreads();
//a[idd]=a[idd]+0.1;
}

c[idd]=swp+0.1; // c[idd]+ny+0.1; // c[idd]+i; // swp+0.1; // c[idd]+(step/steps);
i++; // counts number of writes
if (i > steps-1)
goto nextProcessor; // return;
nl++;
if (i < steps) {
swp=nswp;
}
}
nx++; nl=0;
//if (nx < ux) {
x = (x+dx)%sx; // new position
y = (y+dy)%sy;
z = (z+dz)%sz;
idd=x+sx*y+sx*sy*z;
c[idd]=swp+0.1; // no need to save this value as this is the end of the line
//}
i++;
if (i > steps-1)
goto nextProcessor; // return;
// if (nx <ux) x=(x+1)%sx;
}
ny++;
// if (ny <uy) y=(y+1)%sy;
nx=0;x=0;
}
nz++;
// if (nz <uz) z=(z+1)%sz;
ny=0;y=0;
}
nextProcessor:
nz=0;
}
return;
}