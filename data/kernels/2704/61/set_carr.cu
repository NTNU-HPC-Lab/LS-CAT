#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_carr(float br, float bi, float * c, int N)
{
int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;
int idc=idx*2;
c[idc]=br;c[idc+1]=bi;
}