#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate2(float*a,float b, float * c, int sx,int sy,int sz, int dx, int dy, int dz)
{
int ids=(blockIdx.x*blockDim.x+threadIdx.x); // id of this processor
int x=(ids + dx)%sx;  // advance by the offset steps along the chain
int y=(ids/sx + dy)%sy;
int z=(ids/(sx*sy) + dz)%sz;
int idd=x+sx*y+sx*sy*z;
if(ids>=sx*sy*sz) return;
// float tmp = a[ids];
// __syncthreads();             // nice try but does not work !
c[idd] = b*a[ids];
}