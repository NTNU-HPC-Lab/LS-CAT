#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arr_times_const_checkerboard(float*a,float b, float * c, int N, int sx,int sy,int sz)
{
int ids=blockIdx.x*blockDim.x+threadIdx.x;   // which source array element do I have to deal with?
if(ids>=N) return;  // not in range ... quit

int px=(ids/2)%sx;   // my x pos
int py=(ids/2)/sx;   // my y pos
float minus1=(1-2*((px+py)%2));
c[ids]=a[ids]*b*minus1;
}