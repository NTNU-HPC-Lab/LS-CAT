#include "hip/hip_runtime.h"
#include "includes.h"
/************************* CudaMat ******************************************
*   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
*   heintzmann@gmail.com                                                  *
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; Version 2 of the License.               *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program; if not, write to the                         *
*   Free Software Foundation, Inc.,                                       *
*   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
***************************************************************************
* Compile with:
* Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
* File sudo vi /usr/local/cuda/bin/nvcc.profile
* needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
*/

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177


#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


__global__ void rotate(float*a,float b, float * c, size_t sx,size_t sy,size_t sz, size_t dx, size_t dy, size_t dz, size_t ux, size_t uy, size_t uz)
{
// id of this processor
size_t id=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);

size_t Processes=blockDim.x * gridDim.x;
size_t chains=ux*uy*uz; // total number of independent chains
size_t N=sx*sy*sz;  // total size of array, has to be chains*length_of_chain
size_t length=N/chains;  // chain length
size_t steps=N/Processes;  // this is how many steps each processor has to do

size_t step,nl,nx,ny,nz,x,y,z,i,idd;
float swp, nswp;

//if (id != 0)   return;
//for (id=0;id<Processes;id++)
{
step=steps*id;   // my starting step as the id times the number of steps
nl=step%length;  // current position in chain length
nx=(step/length)%ux;  // current position in unit cell x
ny=(step/(length*ux))%uy;  // current position in unit cell y
nz=(step/(length*ux*uy))%uz;  // current position in unit cell z
i=0;

//if (step/steps != 4 && step/steps != 5) return;

while(nz<uz)
{
while(ny<uy)
{
while (nx<ux)
{
x=(nx+nl*dx)%sx;  // advance by the offset steps along the chain
y=(ny+nl*dy)%sy;
z=(nz+nl*dz)%sz;
idd=x+sx*y+sx*sy*z;
if (i < steps) {
swp=a[idd];
// a[idd]=a[idd]+0.1;
__syncthreads();
}
while (nl<length-1)
{
if (i > steps-1)
goto nextProcessor; // return;
if (step >= N)  // this thread has reached the end of the total data to process
goto nextProcessor; // return;
step++;
x = (x+dx)%sx; // new position
y = (y+dy)%sy;
z = (z+dz)%sz;
idd=x+sx*y+sx*sy*z;
if (i < steps-1) {
nswp=a[idd];
__syncthreads();
//a[idd]=a[idd]+0.1;
}

c[idd]=swp+0.1; // c[idd]+ny+0.1; // c[idd]+i; // swp+0.1; // c[idd]+(step/steps);
i++; // counts number of writes
if (i > steps-1)
goto nextProcessor; // return;
nl++;
if (i < steps) {
swp=nswp;
}
}
nx++; nl=0;
//if (nx < ux) {
x = (x+dx)%sx; // new position
y = (y+dy)%sy;
z = (z+dz)%sz;
idd=x+sx*y+sx*sy*z;
c[idd]=swp+0.1; // no need to save this value as this is the end of the line
//}
i++;
if (i > steps-1)
goto nextProcessor; // return;
// if (nx <ux) x=(x+1)%sx;
}
ny++;
// if (ny <uy) y=(y+1)%sy;
nx=0;x=0;
}
nz++;
// if (nz <uz) z=(z+1)%sz;
ny=0;y=0;
}
nextProcessor:
nz=0;
}
return;
}