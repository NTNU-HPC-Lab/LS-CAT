#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_arr(float b, float * c, int N)
{
int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;
c[idx]=b;
}