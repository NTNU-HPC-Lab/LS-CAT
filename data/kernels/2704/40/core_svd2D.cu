#include "hip/hip_runtime.h"
#include "includes.h"
/************************* CudaMat ******************************************
*   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
*   heintzmann@gmail.com                                                  *
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; Version 2 of the License.               *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program; if not, write to the                         *
*   Free Software Foundation, Inc.,                                       *
*   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
***************************************************************************
* Compile with:
* Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
* File sudo vi /usr/local/cuda/bin/nvcc.profile
* needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
*/

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177


#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


__global__ void core_svd2D(float *X, float *Ye, float * Yv, size_t N){   // N is NOT the total size, but only the size excluding the last dimension (of size 3)
size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);
if(idd>=N) return;
int k;
double n;
double tmp[3];
double E[2];
double U[2];
double trace;
double delta;

for (k=0;k<3;k++)   // get the matrix value [X(1,1) X(2,1)=X(1,2), X(2,2)]
tmp[k]=X[idd+N*k];

if (fabs(tmp[1]) < 1e-15){
E[0]=tmp[0];
E[1]=tmp[2];
U[0]=1.0;
U[1]=0.0;
}
else{
trace=tmp[0]+tmp[2];
delta=(tmp[0]-tmp[2])*(tmp[0]-tmp[2])+4*tmp[1]*tmp[1];
E[0]=0.5*(trace+sqrt(delta));
E[1]=0.5*(trace-sqrt(delta));
n=sqrt((E[0]-tmp[0])*(E[0]-tmp[0])+tmp[1]*tmp[1]);
U[0]=tmp[1]/n;
U[1]=(E[0]-tmp[0])/n;
}

for (k=0;k<2;k++){  // set result
Ye[idd+N*k]=E[k];
Yv[idd+N*k]=U[k];
}
}