#include "hip/hip_runtime.h"
#include "includes.h"
/************************* CudaMat ******************************************
*   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
*   heintzmann@gmail.com                                                  *
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; Version 2 of the License.               *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program; if not, write to the                         *
*   Free Software Foundation, Inc.,                                       *
*   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
***************************************************************************
* Compile with:
* Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
* File sudo vi /usr/local/cuda/bin/nvcc.profile
* needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
*/

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177


#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


__global__ void core_svd2D_recomp(float *Y, float *E, float * V, size_t N){   // N is NOT the total size, but only the size excluding the last dimension (of size 3)
size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);
if(idd>=N) return;
int k;
double ee[2];
double vv[2];
double tmp[3];

for (k=0;k<2;k++){   // get the eigenvalues and eigenvectors
ee[k]=E[idd+N*k];
vv[k]=V[idd+N*k];
}
tmp[0]=ee[0]*pow(vv[0],2) + ee[1]*pow(vv[1],2);
tmp[1]=vv[0]*vv[1]*(ee[0]-ee[1]);
tmp[2]=ee[0]*pow(vv[1],2)+ee[1]*pow(vv[0],2);
for (k=0;k<3;k++){  // set result
Y[idd+N*k]=tmp[k];
}
}