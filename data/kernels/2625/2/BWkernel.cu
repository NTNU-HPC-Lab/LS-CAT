#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
/* 2 bytes de identificación */
uint32_t size;        /* Tamaño del archivo */
uint16_t resv1;       /* Reservado */
uint16_t resv2;       /* Reservado */
uint32_t offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
uint32_t headersize;  /* Tamaño de la cabecera */
uint32_t width;       /* Ancho */
uint32_t height;      /* Alto */
uint16_t planes;      /* Planos de color (Siempre 1) */
uint16_t bpp;         /* bits por pixel */
uint32_t compress;    /* compresion */
uint32_t imgsize;     /* tamaño de los datos de imagen */
uint32_t bpmx;        /* Resolucion X en bits por metro */
uint32_t bpmy;        /* Resolucion Y en bits por metro */
uint32_t colors;      /* colors used en la paleta */
uint32_t imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;


__global__ void BWkernel(unsigned char *img_device, uint32_t n) {
float color;
color = 0.0f;
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) {
color += img_device[i*3 + 0] * 0.114;
color += img_device[i*3 + 1] * 0.587;
color += img_device[i*3 + 2] * 0.299;
color /= 3;
img_device[i*3 + 0] = color;
img_device[i*3 + 1] = color;
img_device[i*3 + 2] = color;
}
}