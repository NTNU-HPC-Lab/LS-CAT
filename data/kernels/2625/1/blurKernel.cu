#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
/* 2 bytes de identificación */
uint32_t size;        /* Tamaño del archivo */
uint16_t resv1;       /* Reservado */
uint16_t resv2;       /* Reservado */
uint32_t offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
uint32_t headersize;  /* Tamaño de la cabecera */
uint32_t width;       /* Ancho */
uint32_t height;      /* Alto */
uint16_t planes;      /* Planos de color (Siempre 1) */
uint16_t bpp;         /* bits por pixel */
uint32_t compress;    /* compresion */
uint32_t imgsize;     /* tamaño de los datos de imagen */
uint32_t bpmx;        /* Resolucion X en bits por metro */
uint32_t bpmy;        /* Resolucion Y en bits por metro */
uint32_t colors;      /* colors used en la paleta */
uint32_t imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;


__global__ void blurKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image) {

int x,y,ile, avgR,avgB,avgG;
int blurSize = 10;
avgB = avgG = avgR = 0;
ile = 0;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int i = width_image * row + col;

if (i < (width_image * height_image)) {
for(x = col; x < width_image && x < col + blurSize; x++)
{
for(y = row; y < height_image && y < row + blurSize; y++)
{
avgB += img_device2[x*3 + y*width_image*3 + 0];
avgG += img_device2[x*3 + y*width_image*3 + 1];
avgR += img_device2[x*3 + y*width_image*3 + 2];
ile++;
}
}
avgB = avgB / ile;
avgG = avgG / ile;
avgR = avgR / ile;

img_device[col*3 + row*width_image*3 + 0] = avgB;
img_device[col*3 + row*width_image*3 + 1] = avgG;
img_device[col*3 + row*width_image*3 + 2] = avgR;
}
}