#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
/* 2 bytes de identificación */
uint32_t size;        /* Tamaño del archivo */
uint16_t resv1;       /* Reservado */
uint16_t resv2;       /* Reservado */
uint32_t offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
uint32_t headersize;  /* Tamaño de la cabecera */
uint32_t width;       /* Ancho */
uint32_t height;      /* Alto */
uint16_t planes;      /* Planos de color (Siempre 1) */
uint16_t bpp;         /* bits por pixel */
uint32_t compress;    /* compresion */
uint32_t imgsize;     /* tamaño de los datos de imagen */
uint32_t bpmx;        /* Resolucion X en bits por metro */
uint32_t bpmy;        /* Resolucion Y en bits por metro */
uint32_t colors;      /* colors used en la paleta */
uint32_t imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;


__global__ void ConvMatKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image, int j, float *mat) {
//Hay que pasarle la matriz
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int i = width_image * row + col;
float avgB, avgG, avgR;
int x, y;

avgB = avgG = avgR = 0;

if (i < (width_image * height_image)) {
for(x = -1; x < 2; x++) {
if (row == 0 && x == -1) {
x = 0;
}
else if (row == height_image - 1) {
if (x > 0) break;
}
for(y = -1; y < 2; y++) {
if (col == 0 && y == -1) y = 0;
if (col == width_image - 1 && y == 1) break;
avgB += img_device[(col + y)*3 + (x + row) * width_image*3 + 0] * mat[((x + 1) * 3) + y + 1];
avgG += img_device[(col + y)*3 + (x + row) * width_image*3 + 1] * mat[((x + 1) * 3) + y + 1];
avgR += img_device[(col + y)*3 + (x + row) * width_image*3 + 2] * mat[((x + 1) * 3) + y + 1];
}
}
img_device2[col*3 + row*width_image*3 + 0] = avgB;
img_device2[col*3 + row*width_image*3 + 1] = avgG;
img_device2[col*3 + row*width_image*3 + 2] = avgR;
}
}