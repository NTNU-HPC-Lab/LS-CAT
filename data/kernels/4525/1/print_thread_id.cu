#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_thread_id()
{
printf("Hello Cuda tid[%d, %d, %d]\n",
threadIdx.x + blockDim.x * blockIdx.x,
threadIdx.y + blockDim.y * blockIdx.y,
threadIdx.z + blockDim.z * blockIdx.z);
//threadIdx.x, threadIdx.y, threadIdx.z);
}