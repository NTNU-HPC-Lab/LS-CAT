#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_thread_variables()
{
printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, BlockDim{%d,%d,%d}, GridDim{%d,%d,%d}\n",
threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z,
blockDim.x, blockDim.y, blockDim.z,
gridDim.x, gridDim.y, gridDim.z
);
}