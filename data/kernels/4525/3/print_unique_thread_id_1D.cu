#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_unique_thread_id_1D()
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;

printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, tid{%d}\n",
threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z,
tid);
}