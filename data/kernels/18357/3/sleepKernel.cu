#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sleepKernel(double* cycles, int64_t waitCycles) {
extern __shared__ int s[];
long long int start = clock64();
for (;;) {
auto total = clock64() - start;
if (total >= waitCycles) { break; }
}
*cycles = (double(clock64() - start));
}