#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unsignedGPU(int numTests, unsigned* ns, unsigned* ds, unsigned* qs, unsigned* rs) {
for (int i = 0; i < numTests; ++i) {
unsigned n = ns[i];
unsigned d = ds[i];

qs[i] = n / d;
rs[i] = n % d;
}
}