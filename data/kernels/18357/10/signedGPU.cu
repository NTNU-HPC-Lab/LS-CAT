#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void signedGPU(int numTests, int* ns, int* ds, int* qs, int* rs) {
for (int i = 0; i < numTests; ++i) {
int n = ns[i];
int d = ds[i];

qs[i] = n / d;
rs[i] = n % d;
}
}