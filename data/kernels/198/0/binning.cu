#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This project is dual licensed. You may license this software under one of the
following licences:

+ Creative Commons Attribution-Share Alike 3.0 Unported License
http://creativecommons.org/licenses/by-nc-sa/3.0/

+ GNU GENERAL PUBLIC LICENSE v3, designated as a "BY-SA Compatible License"
as defined in BY-SA 4.0 on 8 October 2015

* See the LICENSE file in the root directory of this source tree for full
copyright disclosure, and other details.
*/


/* Header files */




/* Constants */

#define threads 256 /* It's the number of threads we are going to use per block on the GPU */

using namespace std;


/* Kernels */

/* This kernel counts the number of pairs in the data file */
/* We will use this kernel to calculate real-real pairs and random-random pairs */


/* This kernel counts the number of pairs that there are between two data groups */
/* We will use this kernel to calculate real-random pairs and real_1-real_2 pairs (cross-correlation) */
/* NOTE that this kernel has NOT been merged with 'binning' above: this is for speed optimization, we avoid passing extra variables to the GPU */


__global__ void binning(float *xd,float *yd,float *zd,float *ZZ,int number_lines,int points_per_degree, int number_of_degrees)
{

/* We define variables (arrays) in shared memory */

float angle;
__shared__ float temp[threads];

/* We define an index to run through these two arrays */

int index = threadIdx.x;

/* This variable is necesary to accelerate the calculation, it's due that "temp" was definied in the shared memory too */

temp[index]=0;
float x,y,z; //MCM
float xx,yy,zz; //MCM

/* We start the counting */

for (int i=0;i<number_lines;i++)
{
x = xd[i];//MCM
y = yd[i];//MCM
z = zd[i];//MCM

/* The "while" replaces the second for-loop in the sequential calculation case (CPU). We use "while" rather than "if" as recommended in the book "Cuda by Example" */

for(int dim_idx = blockIdx.x * blockDim.x + threadIdx.x;
dim_idx < number_lines;
dim_idx += blockDim.x * gridDim.x)
{
xx = xd[dim_idx];//MCM
yy = yd[dim_idx];//MCM
zz = zd[dim_idx];//MCM

/* We make the dot product */
angle = x * xx + y * yy + z * zz;//MCM


//angle[index]=xd[i]*xd[dim_idx]+yd[i]*yd[dim_idx]+zd[i]*zd[dim_idx];//MCM
//__syncthreads();//MCM

/* Sometimes "angle" is higher than one, due to numnerical precision, to solve it we use the next sentence */

angle=fminf(angle,1.0);
angle=acosf(angle)*180.0/M_PI;
//__syncthreads();//MCM

/* We finally count the number of pairs separated an angular distance "angle", always in shared memory */

if(angle < number_of_degrees)
{
atomicAdd( &temp[int(angle*points_per_degree)], 1.0);
}
__syncthreads();
}
}

/* We copy the number of pairs from shared memory to global memory */

atomicAdd( &ZZ[threadIdx.x] , temp[threadIdx.x]);
__syncthreads();
}