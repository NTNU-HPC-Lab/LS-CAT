#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This project is dual licensed. You may license this software under one of the
following licences:

+ Creative Commons Attribution-Share Alike 3.0 Unported License
http://creativecommons.org/licenses/by-nc-sa/3.0/

+ GNU GENERAL PUBLIC LICENSE v3, designated as a "BY-SA Compatible License"
as defined in BY-SA 4.0 on 8 October 2015

* See the LICENSE file in the root directory of this source tree for full
copyright disclosure, and other details.
*/


/* Header files */




/* Constants */

#define threads 256 /* It's the number of threads we are going to use per block on the GPU */

using namespace std;


/* Kernels */

/* This kernel counts the number of pairs in the data file */
/* We will use this kernel to calculate real-real pairs and random-random pairs */


/* This kernel counts the number of pairs that there are between two data groups */
/* We will use this kernel to calculate real-random pairs and real_1-real_2 pairs (cross-correlation) */
/* NOTE that this kernel has NOT been merged with 'binning' above: this is for speed optimization, we avoid passing extra variables to the GPU */


__global__ void binning_mix(float *xd_real, float *yd_real, float *zd_real, float *xd_sim, float *yd_sim, float *zd_sim, float *ZY, int lines_number_1, int lines_number_2, int points_per_degree, int number_of_degrees)
{

/* We define variables (arrays) in shared memory */

float angle;
__shared__ float temp[threads];

/* We define an index to run through these two arrays */

int index = threadIdx.x;

/* This variable is necesary to accelerate the calculation, it's due that "temp" was definied in the shared memory too */

temp[index]=0;
float x,y,z; //MCM
float xx,yy,zz; //MCM

/* We start the counting */

for (int i=0;i<lines_number_1;i++)
{
x = xd_real[i];//MCM
y = yd_real[i];//MCM
z = zd_real[i];//MCM

/* The "while" replaces the second for-loop in the sequential calculation case (CPU). We use "while" rather than "if" as recommended in the book "Cuda by Example" */

for(int dim_idx = blockIdx.x * blockDim.x + threadIdx.x;
dim_idx < lines_number_2;
dim_idx += blockDim.x * gridDim.x)
{
xx = xd_sim[dim_idx];//MCM
yy = yd_sim[dim_idx];//MCM
zz = zd_sim[dim_idx];//MCM
/* We make the dot product */
angle = x * xx + y * yy + z * zz;//MCM

//angle[index]=xd[i]*xd[dim_idx]+yd[i]*yd[dim_idx]+zd[i]*zd[dim_idx];//MCM
//__syncthreads();//MCM

/* Sometimes "angle" is higher than one, due to numnerical precision, to solve it we use the next sentence */

angle=fminf(angle,1.0);
angle=acosf(angle)*180.0/M_PI;
//__syncthreads();//MCM

/* We finally count the number of pairs separated an angular distance "angle", always in shared memory */

if(angle < number_of_degrees)
{
atomicAdd( &temp[int(angle*points_per_degree)], 1.0);
}
__syncthreads();
}
}

/* We copy the number of pairs from shared memory to global memory */

atomicAdd( &ZY[threadIdx.x] , temp[threadIdx.x]);
__syncthreads();
}