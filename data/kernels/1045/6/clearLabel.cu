#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clearLabel(float *prA, float *prB, unsigned int num_nodes, float base)
{
unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
if(id < num_nodes)
{
prA[id] = base + prA[id] * 0.85;
prB[id] = 0;
}
}