#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clearLabel(bool *label, unsigned int size)
{
unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
if(id < size)
label[id] = false;
}