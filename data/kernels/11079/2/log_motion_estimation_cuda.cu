#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void log_motion_estimation_cuda(uint8 *current, uint8 *previous, int *vectors_x, int *vectors_y, int *M_B, int *N_B, int *B, int *M, int *N) {
//obtain idx;
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= ((*M_B) * (*N_B))) return;
int x, y;

x = id / (*M_B);
y = id % (*M_B);


int dd;
for (dd = 4; dd > 1; dd--) {  //--> gives 4 3 2
int step = 0;
if (dd == 4) {
//d=4;
step = 4;
} else if (dd == 3) {
step = 2;
} else if (dd == 2) {
step = 1;
} else {
continue;
}

int min = 255 * (*B) * (*B);
int bestx, besty, i, j, k, l;
for (i = -step; i < step + 1; i += step)      /* For all candidate blocks */
for (j = -step; j < step + 1; j += step) {
int dist = 0;
for (k = 0; k < (*B); k++)        /* For all pixels in the block */
for (l = 0; l < (*B); l++) {
int tmp9 = vectors_x[x * (*M_B) + y];
int p1, p2;
p1 = current[((*B) * x + k) * (*M) + (*B) * y + l];
if (((*B) * x + tmp9 + i + k) < 0 || ((*B) * x + tmp9 + i + k) > ((*N) - 1) ||
((*B) * y + tmp9 + j + l) < 0 || ((*B) * y + tmp9 + j + l) > ((*M) - 1)) {
p2 = 0;
} else {
p2 = previous[((*B) * x + tmp9 + i + k) * (*M) + (*B) * y + tmp9 + j + l];
}

dist += abs(p1 - p2);
}
if (dist < min) {
min = dist;
bestx = i;
besty = j;
}
}

int at = x * (*M_B) + y;

vectors_x[at] += bestx;

vectors_y[at] += besty;

}
}