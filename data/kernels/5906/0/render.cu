#include "hip/hip_runtime.h"
#include "includes.h"



#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ );

__global__ void render( float* framebuffer, int width, int height )
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

if( i >= width || j >= height )
{
return;
}

int pixel_index = j * width * 3 + i * 3;

framebuffer[pixel_index + 0] = float(i) / width;
framebuffer[pixel_index + 1] = float(j) / height;
framebuffer[pixel_index + 2] = 0.2f;
}