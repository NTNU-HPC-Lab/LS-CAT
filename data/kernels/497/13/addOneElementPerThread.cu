#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOneElementPerThread(double* a, double* b, double* c, int n)
{
// Get our global thread ID
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
int id = idy * n + idx;
// Make sure we do not go out of bounds
if (idx < n && idy < n)
c[id] = a[id] + b[id];
}