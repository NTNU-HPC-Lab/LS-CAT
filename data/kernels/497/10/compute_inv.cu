#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_inv(const int* destination_offsets, const int* source_indices, const float* out_degrees, const int node_count, const float* input, float *output)
{
int dest = blockDim.x*blockIdx.x + threadIdx.x;
if (dest<node_count)
{
int srcStart = destination_offsets[dest];
int srcEnd = destination_offsets[dest + 1];
int in_degree = srcEnd - srcStart;
float rank = 0;
if (in_degree>0)
{
for (int srcIdx = srcStart; srcIdx<srcEnd; ++srcIdx)
{
int src = source_indices[srcIdx];
float contrib = ((input[src] * DECAY) * out_degrees[src]);
rank = rank + contrib;
}
}
output[dest] = rank + (1 - DECAY);
}
}