#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_shared(const int* destination_offsets, const int* source_indices, const int* out_degrees, const int node_count, const float* input, float *output)
{
int dest = blockDim.x*blockIdx.x + threadIdx.x;
__shared__ int s_dest_off[BLOCK_SIZE + 1];
if (dest<node_count)
{
s_dest_off[threadIdx.x] = destination_offsets[dest];
if (threadIdx.x == BLOCK_SIZE - 1 || dest == node_count - 1)
{
s_dest_off[threadIdx.x + 1] = destination_offsets[dest + 1];
}
__syncthreads();
int srcStart = s_dest_off[threadIdx.x];
int srcEnd = s_dest_off[threadIdx.x + 1];
int in_degree = srcEnd - srcStart;
float rank = 0;
if (in_degree>0)
{
for (int srcIdx = srcStart; srcIdx<srcEnd; ++srcIdx)
{
int src = source_indices[srcIdx];
float contrib = ((input[src] * DECAY) / out_degrees[src]);
rank = rank + contrib;
}
}
output[dest] = rank + (1 - DECAY);
}
}