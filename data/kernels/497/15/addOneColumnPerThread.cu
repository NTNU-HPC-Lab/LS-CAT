#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOneColumnPerThread(double* a, double* b, double* c, int n)
{
// Get the column for current thread
int column = (blockIdx.x * blockDim.x + threadIdx.x);

// Make sure we do not go out of bounds
if (column < n)
{
for (int i = 0; i < n; i++)
{
c[i * n + column] = a[i * n + column] + b[i * n + column];
}
}
}