#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_abs_diff(float* diff, const float* output1, const float* output2, const int size)
{
extern __shared__ float s_max[];
int i = blockDim.x*blockIdx.x + threadIdx.x;
int tx = threadIdx.x;
if (i<size)
{
float o1 = output1[i];
if (o1 == -1)
{
s_max[tx] = -1;
}
else
{
s_max[tx] = fabsf(o1 - output2[i]);
}
}
else
{
s_max[tx] = -1;
}
for (int j = blockDim.x / 2; j> 0; j >>= 1)
{
__syncthreads();
if (tx<j)
{
s_max[tx] = fmaxf(s_max[tx], s_max[tx + j]);
}
}
if (tx == 0)
{
diff[blockIdx.x] = s_max[0];
}
}