#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(float *g_data, int n)
{
__shared__ float partialSum[NUM_ELEMENTS];
unsigned int t = threadIdx.x;
partialSum[t] = g_data[t];

for (int i = blockDim.x/2; i > 0; i>>=1)
{
__syncthreads();
if(t<i)
{
partialSum[t] += partialSum[t + i];
}
}
if(t==0)
{
g_data[0] = partialSum[0];
}
}