#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void invert(float *output, int* input, const int size)
{
int i = blockDim.x*blockIdx.x + threadIdx.x;
if(i<size)
{
int d = input[i];
if(d>0)
{
output[i] = __fdividef(1.f, d);
}
}
}