#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOneRowPerThread(double* a, double* b, double* c, int n)
{
// Get the row for current thread
int row = (blockIdx.y * blockDim.y + threadIdx.y);

// Make sure we do not go out of bounds
if (row < n)
{
int idx = row * n;
for (int i = 0; i < n; i++)
{
c[idx + i] = a[idx + i] + b[idx + i];
}
}
}