#include "hip/hip_runtime.h"
#include "includes.h"

//Number of elements of the inpu layers, that correspond to the number of pixels of a picture
#define PIXELS 3073
//Number of elements of the first hidden layer
#define HIDDEN_LAYER_1 2000
//Number of elements of the second hidden layer
#define HIDDEN_LAYER_2 450
//Number of elements of the output layer
#define OUTPUT_LAYER 10
//Learning rate of the algorithm
#define LEARNING_RATE 0.01
//Numbers of elements to use for training
#define ELEMENTS 1000
//Blocks
#define BLOCKS 32

/*
* Function that given a vector and its size, print it
* In:
* f: vector of doubles to be printed
* N: size of the vector
*/
__global__ void calculate_error(double *error_vector, double* gradient, double *layer, int N, int M){

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;
if (x < N && y < M)
gradient[x*M+y] = LEARNING_RATE * error_vector[y] * layer[x];

}