#include "hip/hip_runtime.h"
#include "includes.h"

//Number of elements of the inpu layers, that correspond to the number of pixels of a picture
#define PIXELS 3073
//Number of elements of the first hidden layer
#define HIDDEN_LAYER_1 2000
//Number of elements of the second hidden layer
#define HIDDEN_LAYER_2 450
//Number of elements of the output layer
#define OUTPUT_LAYER 10
//Learning rate of the algorithm
#define LEARNING_RATE 0.01
//Numbers of elements to use for training
#define ELEMENTS 1000
//Blocks
#define BLOCKS 32

/*
* Function that given a vector and its size, print it
* In:
* f: vector of doubles to be printed
* N: size of the vector
*/
__global__ void get_layer(double *input, double *matrix, double *result,int input_size, int hidden_size){

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;
if (x < hidden_size && y < input_size)
result[x] += input[y]*matrix[y*hidden_size+x];
}