#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************
* Copyright (c) 2015-2019, ArrayFire
* All rights reserved.
*
* This file is distributed under 3-clause BSD license.
* The complete license agreement can be obtained at:
* http://arrayfire.com/licenses/BSD-3-Clause
********************************************************/

#define USE_FORGE_CUDA_COPY_HELPERS

#define PI 3.14159265359

const unsigned DIMX      = 640;
const unsigned DIMY      = 480;
const float MINIMUM      = 1.0f;
const float MAXIMUM      = 20.f;
const float STEP         = 2.0f;
__global__ void pointGenKernel(float* points, float* dirs, int nelems, float minimum, float step) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

if (i < nelems && j < nelems) {
int id = i + j * nelems;

float x = minimum + i * step;
float y = minimum + j * step;

points[2 * id + 0] = x;
points[2 * id + 1] = y;

dirs[2 * id + 0] = sinf(2.0f * PI * x / 10.f);
dirs[2 * id + 1] = sinf(2.0f * PI * y / 10.f);
}
}