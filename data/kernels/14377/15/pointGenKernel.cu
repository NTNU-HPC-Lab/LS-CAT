#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pointGenKernel(float* points, float* dirs, int nBBS0, int nelems, float minimum, float step) {
int k = blockIdx.x / nBBS0;
int i = blockDim.x * (blockIdx.x - k * nBBS0) + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

if (i < nelems && j < nelems && k < nelems) {
float x = minimum + i * step;
float y = minimum + j * step;
float z = minimum + k * step;

int id = i + j * nelems + k * nelems * nelems;

points[3 * id + 0] = x;
points[3 * id + 1] = y;
points[3 * id + 2] = z;

dirs[3 * id + 0] = x - 10.f;
dirs[3 * id + 1] = y - 10.f;
dirs[3 * id + 2] = z - 10.f;
}
}