#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateCurve(float t, float dx, float* out, const float ZMIN, const size_t ZSIZE) {
int offset = blockIdx.x * blockDim.x + threadIdx.x;

float z = ZMIN + offset * dx;
if (offset < ZSIZE) {
out[3 * offset]     = cos(z * t + t) / z;
out[3 * offset + 1] = sin(z * t + t) / z;
out[3 * offset + 2] = z + 0.1 * sin(t);
}
}