#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_sinf(float* out, const size_t _data_size, int fnCode, const float _dx, const float _frange_start) {
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < _data_size) {
float x  = _frange_start + i * _dx;
int idx  = 2 * i;
out[idx] = x;

switch (fnCode) {
case 0: out[idx + 1] = sinf(x); break;
case 1: out[idx + 1] = cosf(x); break;
case 2: out[idx + 1] = tanf(x); break;
case 3: out[idx + 1] = log10f(x); break;
}
}
}