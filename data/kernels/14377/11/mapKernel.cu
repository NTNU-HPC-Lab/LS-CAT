#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapKernel(float* out, int functionCode, float frange_start, float dx) {
int id  = blockIdx.x * blockDim.x + threadIdx.x;
float x = frange_start + id * dx;
float y;

switch (functionCode) {
case 0: y = cos(x); break;
case 1: y = tan(x); break;
default: y = sin(x); break;
}

out[2 * id + 0] = x;
out[2 * id + 1] = y;
}