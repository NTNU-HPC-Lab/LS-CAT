#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void genColorsKernel(float* colors, int nelems) {
const float AF_BLUE[4]   = {0.0588f, 0.1137f, 0.2745f, 1.0f};
const float AF_ORANGE[4] = {0.8588f, 0.6137f, 0.0745f, 1.0f};

int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < nelems) {
if (i % 2 == 0) {
colors[3 * i + 0] = AF_ORANGE[0];
colors[3 * i + 1] = AF_ORANGE[1];
colors[3 * i + 2] = AF_ORANGE[2];
} else {
colors[3 * i + 0] = AF_BLUE[0];
colors[3 * i + 1] = AF_BLUE[1];
colors[3 * i + 2] = AF_BLUE[2];
}
}
}