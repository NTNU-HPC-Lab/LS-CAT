#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#define TILE 16


/* LU Decomposition using Shared Memory \
\           CUDA                        \
\										\
\ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/


//Initialize a 2D matrix
__global__ void scaleIndex(double *matrix, int n, int index){
int start=(index*n+index);
int end=(index*n+n);

for(int i= start+1 ; i<end; ++i){
matrix[i]=(matrix[i]/matrix[start]);
}

}