#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#define TILE 16


/* LU Decomposition using Shared Memory \
\           CUDA                        \
\										\
\ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/


//Initialize a 2D matrix
__global__ void elim(double *A, int n, int index, int bsize){
extern __shared__ double pivot[];

int idThread=threadIdx.x;
int idBlock=blockIdx.x;
int blockSize=bsize;


if(idThread==0){
for(int i=index;i<n;i++) pivot[i]=A[(index*n)+i];
}

__syncthreads();
//Varitables for pivot, row, start and end
int pivotRow=(index*n);
int currentRow=(((blockSize*idBlock) + idThread)*n);
int start=currentRow+index;
int end=currentRow+n;
//If greater than pivot row, loop from start index + 1(next row) to end of column
if(currentRow >pivotRow){
for(int i= start+1; i<end; ++i){
//Set the matrix value of next row and its column - pivot
A[i]=A[i]-(A[start]*pivot[i-currentRow]);

}
}
}