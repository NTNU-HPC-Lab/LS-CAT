#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
int p = __mul24(y, width) + x;
if (x<width && y<height)
d_Result[p] = d_Data1[p] - d_Data2[p];
__syncthreads();
}