#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addToKPlus(int msize, double* a,  double* b, double* c, double* d)
{
int tid = threadIdx.x; // + blockIdx.x * blockDim.x;
if (tid < msize) {
d[tid] = a[tid] + b[tid] + c[tid];
// tid += blockDim.x*gridDim.x;`
}
}