#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxReduce(int *d_idata, int *d_odata) {
__shared__ int sdata[512];

unsigned int tid = threadIdx.x;
unsigned int index = (blockIdx.x * blockDim.x) + tid;
sdata[tid] = d_idata[index];
__syncthreads();

for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
if (tid < stride){
sdata[tid] = max(sdata[tid], sdata[tid + stride]);
}
}
__syncthreads();

if (tid == 0){
d_odata[blockIdx.x] = sdata[0];
}
}