#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
int j = threadIdx.x + blockDim.x * blockIdx.x;
int i = threadIdx.y + blockDim.y * blockIdx.y;

if (i >= h || j >= w) return;

int flow_row_offset  = i * flow_stride;
int image_row_offset = i * image_stride;

//bottom left corner of a target pixel
float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
// pixel containing bottom left corner
float px;
float py;
float dx = modff (cx, &px);
float dy = modff (cy, &py);
// target pixel integer coords
int tx;
int ty;
tx = (int) px;
ty = (int) py;
float value = src[image_row_offset + j];
float weight;
// fill pixel containing bottom right corner
if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
{
weight = dx * dy;
_atomicAdd (dst + ty * image_stride + tx, value * weight);
_atomicAdd (normalization_factor + ty * image_stride + tx, weight);
}

// fill pixel containing bottom left corner
tx -= 1;
if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
{
weight = (1.0f - dx) * dy;
_atomicAdd (dst + ty * image_stride + tx, value * weight);
_atomicAdd (normalization_factor + ty * image_stride + tx, weight);
}

// fill pixel containing upper left corner
ty -= 1;
if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
{
weight = (1.0f - dx) * (1.0f - dy);
_atomicAdd (dst + ty * image_stride + tx, value * weight);
_atomicAdd (normalization_factor + ty * image_stride + tx, weight);
}

// fill pixel containing upper right corner
tx += 1;
if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
{
weight = dx * (1.0f - dy);
_atomicAdd (dst + ty * image_stride + tx, value * weight);
_atomicAdd (normalization_factor + ty * image_stride + tx, weight);
}
}