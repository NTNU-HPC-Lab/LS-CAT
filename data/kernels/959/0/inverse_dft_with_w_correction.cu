#include "hip/hip_runtime.h"
#include "includes.h"

// Copyright 2019 Adam Campbell, Seth Hall, Andrew Ensor
// Copyright 2019 High Performance Computing Research Laboratory, Auckland University of Technology (AUT)

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from this
// software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.




/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
__device__ double2 complex_multiply(double2 z1, double2 z2)
{
double real = z1.x*z2.x - z1.y*z2.y;
double imag = z1.y*z2.x + z1.x*z2.y;
return make_double2(real, imag);
}
__global__ void inverse_dft_with_w_correction(double2 *grid, size_t grid_pitch, const double3 *visibilities, const double2 *vis_intensity, int vis_count, int batch_count, int x_offset, int y_offset, int render_size, double cell_size)
{
// look up id of thread
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;

if(idx >= render_size || idy >= render_size)
return;

double real_sum = 0;
double imag_sum = 0;

// convert to x and y image coordinates
double x = (idx+x_offset) * cell_size;
double y = (idy+y_offset) * cell_size;

double2 vis;
double2 theta_complex = make_double2(0.0, 0.0);

// precalculate image correction and wCorrection
double image_correction = sqrt(1.0 - (x * x) - (y * y));
double w_correction = image_correction - 1.0;

// NOTE: below is an approximation... Uncomment if needed
// double wCorrection = -((x*x)+(y*y))/2.0;

// loop through all visibilities and create sum using iDFT formula
for(int i = 0; i < batch_count; ++i)
{
double theta = 2.0 * M_PI * (x * visibilities[i].x + y * visibilities[i].y
+ (w_correction * visibilities[i].z));
sincos(theta, &(theta_complex.y), &(theta_complex.x));
vis = complex_multiply(vis_intensity[i], theta_complex);
real_sum += vis.x;
imag_sum += vis.y;
}

// adjust sum by image correction
real_sum *= image_correction;
imag_sum *= image_correction;

// look up destination in image (grid) and divide by amount of visibilities (N)
double2 *row = (double2*)((char*)grid + idy * grid_pitch);
row[idx].x += (real_sum / vis_count);
row[idx].y += (imag_sum / vis_count);
}