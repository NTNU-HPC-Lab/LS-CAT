#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d)
{
return (NX*(NY*(d-1)+y)+x);
}
__global__ void gpu_stream(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2)
{
unsigned int y = blockIdx.y;
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

// streaming step

unsigned int xp1 = (x + 1) % NX;
unsigned int yp1 = (y + 1) % NY;
unsigned int xm1 = (NX + x - 1) % NX;
unsigned int ym1 = (NY + y - 1) % NY;

// direction numbering scheme
// 6 2 5
// 3 0 1
// 7 4 8

// load populations from adjacent nodes (ft is post-streaming population of f1)
f1[gpu_fieldn_index(x, y, 1)] = f2[gpu_fieldn_index(xm1, y, 1)];
f1[gpu_fieldn_index(x, y, 2)] = f2[gpu_fieldn_index(x, ym1, 2)];
f1[gpu_fieldn_index(x, y, 3)] = f2[gpu_fieldn_index(xp1, y, 3)];
f1[gpu_fieldn_index(x, y, 4)] = f2[gpu_fieldn_index(x, yp1, 4)];
f1[gpu_fieldn_index(x, y, 5)] = f2[gpu_fieldn_index(xm1, ym1, 5)];
f1[gpu_fieldn_index(x, y, 6)] = f2[gpu_fieldn_index(xp1, ym1, 6)];
f1[gpu_fieldn_index(x, y, 7)] = f2[gpu_fieldn_index(xp1, yp1, 7)];
f1[gpu_fieldn_index(x, y, 8)] = f2[gpu_fieldn_index(xm1, yp1, 8)];

h1[gpu_fieldn_index(x, y, 1)] = h2[gpu_fieldn_index(xm1, y, 1)];
h1[gpu_fieldn_index(x, y, 2)] = h2[gpu_fieldn_index(x, ym1, 2)];
h1[gpu_fieldn_index(x, y, 3)] = h2[gpu_fieldn_index(xp1, y, 3)];
h1[gpu_fieldn_index(x, y, 4)] = h2[gpu_fieldn_index(x, yp1, 4)];
h1[gpu_fieldn_index(x, y, 5)] = h2[gpu_fieldn_index(xm1, ym1, 5)];
h1[gpu_fieldn_index(x, y, 6)] = h2[gpu_fieldn_index(xp1, ym1, 6)];
h1[gpu_fieldn_index(x, y, 7)] = h2[gpu_fieldn_index(xp1, yp1, 7)];
h1[gpu_fieldn_index(x, y, 8)] = h2[gpu_fieldn_index(xm1, yp1, 8)];
}