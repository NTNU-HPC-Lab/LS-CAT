#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y)
{
return NX*y+x;
}
__global__ void gpu_efield(double *fi, double *ex, double *ey){

unsigned int y = blockIdx.y;
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int xp1 = (x + 1) % NX;
unsigned int yp1 = (y + 1) % NY;
unsigned int xm1 = (NX + x - 1) % NX;
unsigned int ym1 = (NY + y - 1) % NY;
double phi  = fi[gpu_scalar_index(x, y)];
double phiL = fi[gpu_scalar_index(xm1, y)];
double phiR = fi[gpu_scalar_index(xp1, y)];
double phiU = fi[gpu_scalar_index(x, yp1)];
double phiD = fi[gpu_scalar_index(x, ym1)];
ex[gpu_scalar_index(x, y)] = 0.5*(phiL - phiR) / dx;
ey[gpu_scalar_index(x, y)] = 0.5*(phiD - phiU) / dy;
}