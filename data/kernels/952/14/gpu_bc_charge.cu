#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d)
{
return (NX*(NY*(d-1)+y)+x);
}
__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y)
{
return NX*y+x;
}
__global__ void gpu_bc_charge(double *h0, double *h1, double *h2)
{
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y;

perturb = 0;

if (y == 0) {
double multi0c = 2.0*charge0*w0;
double multisc = 2.0*charge0*ws;
double multidc = 2.0*charge0*wd;
// lower plate for charge density

double ht1 = h2[gpu_fieldn_index(x, 0, 1)];
double ht2 = h2[gpu_fieldn_index(x, 0, 2)];
double ht3 = h2[gpu_fieldn_index(x, 0, 3)];
double ht4 = h2[gpu_fieldn_index(x, 0, 4)];
double ht5 = h2[gpu_fieldn_index(x, 0, 5)];
double ht6 = h2[gpu_fieldn_index(x, 0, 6)];
double ht7 = h2[gpu_fieldn_index(x, 0, 7)];
double ht8 = h2[gpu_fieldn_index(x, 0, 8)];
// lower plate for constant charge density

h0[gpu_field0_index(x, 0)] = -h0[gpu_field0_index(x, 0)] + multi0c;
h1[gpu_fieldn_index(x, 0, 3)] = -ht1 + multisc;
h1[gpu_fieldn_index(x, 0, 4)] = -ht2 + multisc;
h1[gpu_fieldn_index(x, 0, 1)] = -ht3 + multisc;
h1[gpu_fieldn_index(x, 0, 2)] = -ht4 + multisc;
h1[gpu_fieldn_index(x, 0, 7)] = -ht5 + multidc;
h1[gpu_fieldn_index(x, 0, 8)] = -ht6 + multidc;
h1[gpu_fieldn_index(x, 0, 5)] = -ht7 + multidc;
h1[gpu_fieldn_index(x, 0, 6)] = -ht8 + multidc;
}
}