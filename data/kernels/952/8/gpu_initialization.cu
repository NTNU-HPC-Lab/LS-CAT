#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y)
{
return NX*y+x;
}
__global__ void gpu_initialization(double *r, double *c, double *fi, double *u, double *v, double *ex, double *ey)
{
unsigned int y = blockIdx.y;
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
size_t sidx = gpu_scalar_index(x, y);
r[sidx]  = rho0;
c[sidx]  = 0.0;
fi[sidx] = voltage * (Ly - dy*y) / Ly;
u[sidx]  = 0.0;
v[sidx]  = 0.0;
ex[sidx] = 0.0;
ey[sidx] = 0.0;
}