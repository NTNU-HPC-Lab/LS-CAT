#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y)
{
return NX*y+x;
}
__device__ __forceinline__ size_t gpu_s_scalar_index(unsigned int x, unsigned int y)
{
return (2*RAD + nThreads)*y + x;
}
__global__ void gpu_poisson(double *c, double *fi,double *R){
unsigned int y   = blockIdx.y;
unsigned int x   = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int s_y = threadIdx.y + RAD;
unsigned int s_x = threadIdx.x + RAD;
unsigned int xp1 = (x + blockDim.x) % NX;
unsigned int yp1 = (y + blockDim.y) % NY;
unsigned int xm1 = (NX + x - 1) % NX;
unsigned int ym1 = (NY + y - 1) % NY;
__shared__ double s_in[(2*RAD + nThreads)*3];
// load to shared memory (regular cells)
s_in[gpu_s_scalar_index(s_x,s_y)] = fi[gpu_scalar_index(x, y)];

// load halo cells
if (threadIdx.x < RAD) {
s_in[gpu_s_scalar_index(s_x - RAD, s_y)] = fi[gpu_scalar_index(xm1, y)];
s_in[gpu_s_scalar_index(s_x + blockDim.x, s_y)] = fi[gpu_scalar_index(xp1, y)];
}
if (threadIdx.y < RAD) {
s_in[gpu_s_scalar_index(s_x, s_y - RAD)] = fi[gpu_scalar_index(x, ym1)];
s_in[gpu_s_scalar_index(s_x, s_y + blockDim.y)] = fi[gpu_scalar_index(x, yp1)];
}
// Boundary conditions
if (y == 0) {
fi[gpu_scalar_index(x, y)] = voltage;
return;
}
if (y == NY - 1) {
fi[gpu_scalar_index(x, y)] = 0.0;
return;
}
__syncthreads();

double charge    = c[gpu_scalar_index(x, y)];
//double phi       = fi[gpu_scalar_index(x, y)];
//double phiL      = fi[gpu_scalar_index(xm1, y)];
//double phiR      = fi[gpu_scalar_index(xp1, y)];
//double phiU      = fi[gpu_scalar_index(x, yp1)];
//double phiD      = fi[gpu_scalar_index(x, ym1)];

double phi  = s_in[gpu_s_scalar_index(s_x, s_y)];
double phiL = s_in[gpu_s_scalar_index(s_x-1, s_y)];
double phiR = s_in[gpu_s_scalar_index(s_x+1, s_y)];
double phiU = s_in[gpu_s_scalar_index(s_x, s_y+1)];
double phiD = s_in[gpu_s_scalar_index(s_x, s_y-1)];

double source    = (charge / eps) * dx *dx; // Right hand side of the equation
double phi_old   = phi;
phi = 0.25 * (phiL + phiR + phiU + phiD + source);
// Record the error
R[gpu_scalar_index(x, y)] = fabs(phi - phi_old);

//__syncthreads();
fi[gpu_scalar_index(x, y)] = phi;
//if (x == 5 && y == 5) printf("%g\n", phi);
}