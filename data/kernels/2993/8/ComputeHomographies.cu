#include "hip/hip_runtime.h"
#include "includes.h"

//================= Device matching functions =====================//




// Version based on suggestion by Nicholas Lin

#define FMC2W 16
#define FMC2H 4





__device__ volatile int lock = 0;



template <int size>
__device__ void InvertMatrix(float elem[size][size], float res[size][size])
{
int indx[size];
float b[size];
float vv[size];
for (int i=0;i<size;i++)
indx[i] = 0;
int imax = 0;
float d = 1.0;
for (int i=0;i<size;i++) { // find biggest element for each row
float big = 0.0;
for (int j=0;j<size;j++) {
float temp = fabs(elem[i][j]);
if (temp>big)
big = temp;
}
if (big>0.0)
vv[i] = 1.0/big;
else
vv[i] = 1e16;
}
for (int j=0;j<size;j++) {
for (int i=0;i<j;i++) { // i<j
float sum = elem[i][j]; // i<j (lower left)
for (int k=0;k<i;k++) // k<i<j
sum -= elem[i][k]*elem[k][j]; // i>k (upper right), k<j (lower left)
elem[i][j] = sum; // i<j (lower left)
}
float big = 0.0;
for (int i=j;i<size;i++) { // i>=j
float sum = elem[i][j]; // i>=j (upper right)
for (int k=0;k<j;k++) // k<j<=i
sum -= elem[i][k]*elem[k][j]; // i>k (upper right), k<j (lower left)
elem[i][j] = sum; // i>=j (upper right)
float dum = vv[i]*fabs(sum);
if (dum>=big) {
big = dum;
imax = i;
}
}
if (j!=imax) { // imax>j
for (int k=0;k<size;k++) {
float dum = elem[imax][k]; // upper right and lower left
elem[imax][k] = elem[j][k];
elem[j][k] = dum;
}
d = -d;
vv[imax] = vv[j];
}
indx[j] = imax;
if (elem[j][j]==0.0)  // j==j (upper right)
elem[j][j] = 1e-16;
if (j!=(size-1)) {
float dum = 1.0/elem[j][j];
for (int i=j+1;i<size;i++) // i>j
elem[i][j] *= dum; // i>j (upper right)
}
}
for (int j=0;j<size;j++) {
for (int k=0;k<size;k++)
b[k] = 0.0;
b[j] = 1.0;
int ii = -1;
for (int i=0;i<size;i++) {
int ip = indx[i];
float sum = b[ip];
b[ip] = b[i];
if (ii!=-1)
for (int j=ii;j<i;j++)
sum -= elem[i][j]*b[j]; // i>j (upper right)
else if (sum!=0.0)
ii = i;
b[i] = sum;
}
for (int i=size-1;i>=0;i--) {
float sum = b[i];
for (int j=i+1;j<size;j++)
sum -= elem[i][j]*b[j]; // i<j (lower left)
b[i] = sum/elem[i][i]; // i==i (upper right)
}
for (int i=0;i<size;i++)
res[i][j] = b[i];
}
}
__global__ void ComputeHomographies(float *coord, int *randPts, float *homo, int numPts)
{
float a[8][8], ia[8][8];
float b[8];
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int idx = blockDim.x*bx + tx;
const int numLoops = blockDim.x*gridDim.x;
for (int i=0;i<4;i++) {
int pt = randPts[i*numLoops+idx];
float x1 = coord[pt+0*numPts];
float y1 = coord[pt+1*numPts];
float x2 = coord[pt+2*numPts];
float y2 = coord[pt+3*numPts];
float *row1 = a[2*i+0];
row1[0] = x1;
row1[1] = y1;
row1[2] = 1.0;
row1[3] = row1[4] = row1[5] = 0.0;
row1[6] = -x2*x1;
row1[7] = -x2*y1;
float *row2 = a[2*i+1];
row2[0] = row2[1] = row2[2] = 0.0;
row2[3] = x1;
row2[4] = y1;
row2[5] = 1.0;
row2[6] = -y2*x1;
row2[7] = -y2*y1;
b[2*i+0] = x2;
b[2*i+1] = y2;
}
InvertMatrix<8>(a, ia);
__syncthreads();
for (int j=0;j<8;j++) {
float sum = 0.0f;
for (int i=0;i<8;i++)
sum += ia[j][i]*b[i];
homo[j*numLoops+idx] = sum;
}
__syncthreads();
}