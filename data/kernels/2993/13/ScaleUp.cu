#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
#undef BW
#undef BH
#define BW (SCALEUP_W/2 + 2)
#define BH (SCALEUP_H/2 + 2)
__shared__ float buffer[BW*BH];
const int tx = threadIdx.x;
const int ty = threadIdx.y;
if (tx<BW && ty<BH) {
int x = min(max(blockIdx.x*(SCALEUP_W/2) + tx - 1, 0), width-1);
int y = min(max(blockIdx.y*(SCALEUP_H/2) + ty - 1, 0), height-1);
buffer[ty*BW + tx] = d_Data[y*pitch + x];
}
__syncthreads();
int x = blockIdx.x*SCALEUP_W + tx;
int y = blockIdx.y*SCALEUP_H + ty;
if (x<2*width && y<2*height) {
int bx = (tx + 1)/2;
int by = (ty + 1)/2;
int bp = by*BW + bx;
float wx = 0.25f + (tx&1)*0.50f;
float wy = 0.25f + (ty&1)*0.50f;
d_Result[y*newpitch + x] = wy*(wx*buffer[bp] + (1.0f-wx)*buffer[bp+1]) +
(1.0f-wy)*(wx*buffer[bp+BW] + (1.0f-wx)*buffer[bp+BW+1]);
}
}