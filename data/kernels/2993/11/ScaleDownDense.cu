#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ScaleDownDense(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
#define BW (SCALEDOWN_W+4)
#define BH (SCALEDOWN_H+4)
#define W2 (SCALEDOWN_W/2)
#define H2 (SCALEDOWN_H/2)
__shared__ float irows[BH*BW];
__shared__ float brows[BH*W2];
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int xp = blockIdx.x*SCALEDOWN_W + tx;
const int yp = blockIdx.y*SCALEDOWN_H + ty;
const int xl = min(width-1,  max(0, xp-2));
const int yl = min(height-1, max(0, yp-2));
const float k0 = d_ScaleDownKernel[0];
const float k1 = d_ScaleDownKernel[1];
const float k2 = d_ScaleDownKernel[2];
if (xp<(width+4) && yp<(height+4))
irows[BW*ty + tx] = d_Data[yl*pitch + xl];
__syncthreads();
if (yp<(height+4) && tx<W2) {
float *ptr = &irows[BW*ty + 2*tx];
brows[W2*ty + tx] = k0*(ptr[0] + ptr[4]) + k1*(ptr[1] + ptr[3]) + k2*ptr[2];
}
__syncthreads();
const int xs = blockIdx.x*W2 + tx;
const int ys = blockIdx.y*H2 + ty;
if (tx<W2 && ty<H2 && xs<(width/2) && ys<(height/2)) {
float *ptr = &brows[W2*(ty*2) + tx];
d_Result[ys*newpitch + xs] = k0*(ptr[0] + ptr[4*W2]) + k1*(ptr[1*W2] + ptr[3*W2]) + k2*ptr[2*W2];
}
}