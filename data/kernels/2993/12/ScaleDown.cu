#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
__shared__ float inrow[SCALEDOWN_W+4];
__shared__ float brow[5*(SCALEDOWN_W/2)];
__shared__ int yRead[SCALEDOWN_H+4];
__shared__ int yWrite[SCALEDOWN_H+4];
#define dx2 (SCALEDOWN_W/2)
const int tx = threadIdx.x;
const int tx0 = tx + 0*dx2;
const int tx1 = tx + 1*dx2;
const int tx2 = tx + 2*dx2;
const int tx3 = tx + 3*dx2;
const int tx4 = tx + 4*dx2;
const int xStart = blockIdx.x*SCALEDOWN_W;
const int yStart = blockIdx.y*SCALEDOWN_H;
const int xWrite = xStart/2 + tx;
float k0 = d_ScaleDownKernel[0];
float k1 = d_ScaleDownKernel[1];
float k2 = d_ScaleDownKernel[2];
if (tx<SCALEDOWN_H+4) {
int y = yStart + tx - 2;
y = (y<0 ? 0 : y);
y = (y>=height ? height-1 : y);
yRead[tx] = y*pitch;
yWrite[tx] = (yStart + tx - 4)/2 * newpitch;
}
__syncthreads();
int xRead = xStart + tx - 2;
xRead = (xRead<0 ? 0 : xRead);
xRead = (xRead>=width ? width-1 : xRead);

int maxtx = min(dx2, width/2 - xStart/2);
for (int dy=0;dy<SCALEDOWN_H+4;dy+=5) {
{
inrow[tx] = d_Data[yRead[dy+0] + xRead];
__syncthreads();
if (tx<maxtx) {
brow[tx4] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
if (dy>=4 && !(dy&1))
d_Result[yWrite[dy+0] + xWrite] = k2*brow[tx2] + k0*(brow[tx0]+brow[tx4]) + k1*(brow[tx1]+brow[tx3]);
}
__syncthreads();
}
if (dy<(SCALEDOWN_H+3)) {
inrow[tx] = d_Data[yRead[dy+1] + xRead];
__syncthreads();
if (tx<maxtx) {
brow[tx0] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
if (dy>=3 && (dy&1))
d_Result[yWrite[dy+1] + xWrite] = k2*brow[tx3] + k0*(brow[tx1]+brow[tx0]) + k1*(brow[tx2]+brow[tx4]);
}
__syncthreads();
}
if (dy<(SCALEDOWN_H+2)) {
inrow[tx] = d_Data[yRead[dy+2] + xRead];
__syncthreads();
if (tx<maxtx) {
brow[tx1] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
if (dy>=2 && !(dy&1))
d_Result[yWrite[dy+2] + xWrite] = k2*brow[tx4] + k0*(brow[tx2]+brow[tx1]) + k1*(brow[tx3]+brow[tx0]);
}
__syncthreads();
}
if (dy<(SCALEDOWN_H+1)) {
inrow[tx] = d_Data[yRead[dy+3] + xRead];
__syncthreads();
if (tx<maxtx) {
brow[tx2] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
if (dy>=1 && (dy&1))
d_Result[yWrite[dy+3] + xWrite] = k2*brow[tx0] + k0*(brow[tx3]+brow[tx2]) + k1*(brow[tx4]+brow[tx1]);
}
__syncthreads();
}
if (dy<SCALEDOWN_H) {
inrow[tx] = d_Data[yRead[dy+4] + xRead];
__syncthreads();
if (tx<dx2 && xWrite<width/2) {
brow[tx3] = k0*(inrow[2*tx]+inrow[2*tx+4]) + k1*(inrow[2*tx+1]+inrow[2*tx+3]) + k2*inrow[2*tx+2];
if (!(dy&1))
d_Result[yWrite[dy+4] + xWrite] = k2*brow[tx1] + k0*(brow[tx4]+brow[tx3]) + k1*(brow[tx0]+brow[tx2]);
}
__syncthreads();
}
}
}