#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void k_count_received(int nr_total_blocks, uint* d_n_recv_by_block, uint* d_spine_cnts)
{
int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

if (bid < nr_total_blocks) {
d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = d_n_recv_by_block[bid];
}
}