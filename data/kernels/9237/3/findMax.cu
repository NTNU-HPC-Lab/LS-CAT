#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findMax(int *m, int *cs, int n)
{
// your code goes here
int colnum = blockDim.x * blockIdx.x + threadIdx.x;
int max = m[0];
for (int    k = 0; k < n; k++){
if(m [colnum+n*k] > max)
max = m [colnum+n*k];
}
cs[colnum] = max;
}