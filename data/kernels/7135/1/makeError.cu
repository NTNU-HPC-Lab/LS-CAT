#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;// find specific index/thread in GPU
const int size = blockDim.x * gridDim.x; // the size of all index/thread in GPU

for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
}
}