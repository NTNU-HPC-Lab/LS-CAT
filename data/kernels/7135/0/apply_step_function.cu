#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float step_function(float v) //Sigmoid function::Activation Function
{
return 1 / (1 + exp(-v));
}
__global__ void apply_step_function(float *input, float *output, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int size = blockDim.x * gridDim.x;

for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
output[idx] = step_function(input[idx]);
}
}