#include "hip/hip_runtime.h"
#include "includes.h"


__shared__ int smem[324];

__global__ void convolution1Kernel(int *dst, int *src, int rows, int cols, int *filter) {
// Convolucion en memoria global, similar a la convolucion en CPU
int posx = threadIdx.x + blockIdx.x * blockDim.x;
int posy = threadIdx.y + blockIdx.y * blockDim.y;
if (posx > 0 && posy > 0 && posx < rows - 1 && posy < cols - 1) {
for (int k = 0; k < 3; ++k) {
for (int l = 0; l < 3; ++l) {
dst[posy * cols + posx] += src[(posy + k - 1) * cols + (posx + l - 1)] * filter[k * 3 + l];
//printf("Fuente = %i \n", src[(posy + k - 1) * cols + (posx + l - 1)]);
//printf("Filtro = %i \n", filter[k * 3 + l]);

}
}
}
//printf("Destino = %i \n", dst[posy * cols + posx]);

}