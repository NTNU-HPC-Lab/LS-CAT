#include "hip/hip_runtime.h"
#include "includes.h"


#define MAXN 8000  /* Max value of N */
int N;  /* Matrix Dimension*/
int numThreads;  /* Number of Threads */

/*Random*/
#define randm() 4|2[uid]&3

/*CUDA Function for calculating mean column-wise and then reducing each column's totals*/
/*This Function will be called Number of blocks times*/

/* returns a seed for srand based on the time */
__global__ void Mean_SD_Norm(float* input,float* output ,float* mean_out,float* sd_out, int dim1, int numThread,int eval_ceil)
{
extern __shared__ float mean[];//shared 1D-matrix for storing temporary results for mean of each threads
extern __shared__ float sd[];//shared 1D-matrix for storing temporary results for sd of each threads
__shared__ float meansum;//shared 1D-matrix for storing mean total of each threads
__shared__ float sdsum;//shared 1D-matrix for storing SD total of each threads

int idx_x = blockIdx.x * blockDim.x + threadIdx.x;//Getting Thread X Index for Particular Block
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;//Getting Thread Y Index for Particular Block
int eva_block,index;

unsigned int thread_id = threadIdx.y;//Getting Id of thread
unsigned int j = idx_y * dim1 + idx_x;//calculating index for input matrix

__syncthreads();//waiting for all threads

mean[thread_id]=input[j];//Assigned each column element of matrix to each thread

/*If Dimension is more than Threads then reduce the remaining elements to assigned elements*/
for(int i=0;i<dim1;i+=numThread)
{
index=dim1*(numThread+thread_id+i);//calculating index of remaining element
eva_block=index+blockIdx.x;
if(eva_block < dim1*dim1)
{
mean[thread_id]+=input[index];
}
}

/*Reducing sum of each thread to final block sum*/
if(thread_id==0)
{
for(int i=0;i<numThread;i++)
{
meansum+=mean[thread_id+i];
}
mean_out[blockIdx.x]=meansum/dim1;//Mean of block
}

__syncthreads();
sd[thread_id] = powf(input[j] - mean_out[blockIdx.x], 2.0);//evaluating SD for each thread for particular block


/*If Dimension is more than Threads then reduce the remaining elements to assigned elements*/
for(int i=0;i<dim1;i+=numThread)
{
index=dim1*(numThread+thread_id+i);
eva_block=index+blockIdx.x;
if(eva_block < dim1*dim1)
{
sd[thread_id]+=powf(input[index] - mean_out[blockIdx.x], 2.0);
}
}

/*Reducing SD Sum of each thread to final block SD sum*/
if(thread_id==0)
{
sdsum=0;
for(int i=0;i<numThread;i++)
{
sdsum+=sd[thread_id+i];//calculating index of remaining element
}
sd_out[blockIdx.x]=sdsum/dim1;//SD of block
}

__syncthreads();//waiting for threads

/*Normalization of each block data on basis of mean and sd of each block*/
output[blockIdx.x*dim1+thread_id] = (input[thread_id+blockIdx.x*dim1] - mean_out[blockIdx.x]) / sd_out[blockIdx.x];

/*Reducing Normalized Sum for remaining elements*/
for(int i=0;i<eval_ceil;i++){
if((numThread+thread_id)+blockIdx.x*dim1 < dim1*dim1)
{
output[(numThread+thread_id)+blockIdx.x*dim1] = (input[(numThread+thread_id)+blockIdx.x*dim1] - mean_out[blockIdx.x])/sd_out[blockIdx.x];//Normalizing the Matrix Indexes
}
}
}