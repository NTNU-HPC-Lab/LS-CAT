#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "im2col_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int n = XSIZE*YSIZE;
float *data_im = NULL;
hipMalloc(&data_im, XSIZE*YSIZE);
int height = YSIZE;
int width = XSIZE;
int ksize_h = XSIZE*YSIZE;
int ksize_w = XSIZE*YSIZE;
int pad_h = 1;
int pad_w = 1;
int stride_h = 2;
int stride_w = 2;
int dilation_h = 1;
int dilation_w = 1;
int height_col = YSIZE;
int width_col = XSIZE;
float *data_col = NULL;
hipMalloc(&data_col, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
im2col_kernel<<<gridBlock,threadBlock>>>(n,data_im,height,width,ksize_h,ksize_w,pad_h,pad_w,stride_h,stride_w,dilation_h,dilation_w,height_col,width_col,data_col);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
im2col_kernel<<<gridBlock,threadBlock>>>(n,data_im,height,width,ksize_h,ksize_w,pad_h,pad_w,stride_h,stride_w,dilation_h,dilation_w,height_col,width_col,data_col);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
im2col_kernel<<<gridBlock,threadBlock>>>(n,data_im,height,width,ksize_h,ksize_w,pad_h,pad_w,stride_h,stride_w,dilation_h,dilation_w,height_col,width_col,data_col);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}