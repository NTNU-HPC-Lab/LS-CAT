#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_kernel(int n, float* data_im, int height, int width, int ksize_h, int ksize_w, int pad_h, int pad_w, int stride_h, int stride_w, int dilation_h, int dilation_w, int height_col, int width_col, float* data_col) {
for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (n); index += blockDim.x * gridDim.x) {
int w_out = index % width_col;
index /= width_col;
int h_out = index % height_col;
int channel_in = index / height_col;
int channel_out = channel_in * ksize_h * ksize_w;
int h_in = h_out * stride_h - pad_h;
int w_in = w_out * stride_w - pad_w;
data_col += (channel_out * height_col + h_out) * width_col + w_out;
data_im += (channel_in * height + h_in) * width + w_in;
for (int i = 0; i < ksize_h; ++i) {
for (int j = 0; j < ksize_w; ++j) {
int h = h_in + i * dilation_h;
int w = w_in + j * dilation_w;
*data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
data_im[i * dilation_h * width + j * dilation_w] : 0;
data_col += height_col * width_col;
}
}
}
}