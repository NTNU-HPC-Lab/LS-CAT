#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void encode_cols_kernel(float *a, uint32_t* b, int m, int n) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;
int i32 = i*ENCODE_BITS;
if (j < n && i32 < m) {
uint32_t r = 0;
for(int k = 0; j + n * (i32 + k)< m * n && k < ENCODE_BITS; k++){
r |= (a[j + n * (i32 + k)]>0)<<k;
}
b[j + n * i] = r;
}
}