#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binary_gemm_kernel(uint32_t* A, uint32_t* B, float* C, int m, int nn, int k, int transb, int alpha, int beta, float *alphas) {
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

int row = threadIdx.y;
int col = threadIdx.x;

int n = 1 + (nn-1)/ENCODE_BITS;
int startLocation = BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol;

float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

__shared__ uint32_t As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ uint32_t Bs[BLOCK_SIZE][BLOCK_SIZE];

int Cvalue = 0;

int c = blockIdx.x*blockDim.x + threadIdx.x;
int r = blockIdx.y*blockDim.y + threadIdx.y;
int lim = 1+( (n-1) / BLOCK_SIZE);
for (int i = 0; i < lim; ++i) {

// Get sub-matrix Asub of A
uint32_t* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];

// Get sub-matrix Bsub of B
uint32_t* Bsub = transb? &B[BLOCK_SIZE * blockCol * n + BLOCK_SIZE * i] : &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];

if ((BLOCK_SIZE*i+col)<n && r<m)
As[row][col] = Asub[row*n+col];
else
As[row][col] = 0;
if ((BLOCK_SIZE*i+row)<n && c<k)
Bs[row][col] = transb? Bsub[row+col*n] : Bsub[row*k+col];
else
Bs[row][col] = 0;

__syncthreads();
#pragma unroll
for (int j = 0; j < BLOCK_SIZE; ++j)
Cvalue += __popc(As[row][j]^Bs[j][col]);
__syncthreads();
}

if(col + blockCol* BLOCK_SIZE< k && row + blockRow* BLOCK_SIZE< m){
Csub[row*k+col] = beta ? Csub[row*k+col]:0;
Csub[row*k+col]+= alpha? (1.0*nn-(Cvalue<<1))*alphas[(startLocation+row*k+col)/k] : 1.0*nn-(Cvalue<<1);
}
}