#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cpy(int *a, int *b, int n) {
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
while (i < n) {
int val = b[i];
sum += val;
i += blockDim.x * gridDim.x;
}
atomicAdd(a, sum);
}