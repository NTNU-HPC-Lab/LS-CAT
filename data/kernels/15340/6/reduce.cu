#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *a, int *b, int n) {
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
atomicAdd(b, a[i]);
}