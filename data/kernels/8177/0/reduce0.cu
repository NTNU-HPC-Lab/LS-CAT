#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;


// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c



__global__ void reduce0(float* g_odata, float* g_idata1, float* g_idata2) {
extern __shared__ float sdata[];
// each thread loads one element from global to shared mem

unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
sdata[tid] = g_idata1[i] * g_idata2[i];
__syncthreads();
// do reduction in shared mem
for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}
// write result for this block to global mem
if (tid == 0) {
g_odata[blockIdx.x] = sdata[0];
//atomicAdd(g_odata, sdata[0]);
}
}