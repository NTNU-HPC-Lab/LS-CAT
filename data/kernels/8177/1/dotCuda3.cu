#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;


// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c



__global__ void dotCuda3(float *a, float *b, float *c){
__shared__ float cache[1024];
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int cacheIndex = threadIdx.x;
float temp = a[tid] * b[tid];//+ a[tid + blockDim.x] * b[tid + blockDim.x];
cache[cacheIndex] = temp;
__syncthreads();

for (unsigned int i = blockDim.x >> 1; i > 0; i >>= 1) {
if (cacheIndex < i)
cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
}

if (cacheIndex == 0){
c[blockIdx.x] = cache[0];
}
}