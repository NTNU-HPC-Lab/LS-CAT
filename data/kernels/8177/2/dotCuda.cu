#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;


// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c



__global__ void dotCuda(float* tmp, float* t1, float* t2, int size) {
//unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

tmp[i] = t1[i] * t2[i];
__syncthreads();

int mididx = size / 2;

while (i < mididx) {
tmp[i] += tmp[i + mididx];
mididx /= 2;
__syncthreads();
}
//atomicAdd(tmp, p);
}