#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;


// https://stackoverflow.com/questions/26853363/dot-product-for-dummies-with-cuda-c



__global__ void init_vec(float* vec, float value) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
vec[tid] = value;
}