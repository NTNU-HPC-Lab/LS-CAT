#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
int thread_id = threadIdx.x;
int block_id = blockIdx.x;
float *Ar = A_in + block_id * ldi;
float *Aw = A_out + block_id * ldo;

int half_length = row_length / 2;
for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
// ignore special case
if (idx == 0) continue;

float2 val = reinterpret_cast<float2 *>(Ar)[idx];
float ret = val.x * val.x + val.y * val.y;
if (use_power) {
Aw[idx] = ret;
} else {
Aw[idx] = sqrtf(ret);
}
}

// handle special case
if (threadIdx.x == 0) {
float real = Ar[0];
// cufft puts this at the end, this is different than kaldi does with its
// own
// internal implementation
float im = Ar[row_length];

if (use_power) {
Aw[0] = real * real;
Aw[half_length] = im * im;
} else {
Aw[0] = fabs(real);
Aw[half_length] = fabs(im);
}
}
}