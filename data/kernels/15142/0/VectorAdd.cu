#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE_thread 1024



__global__ void VectorAdd(int *A, int *B, int *C,int n)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
if(i<n)
C[i]=A[i]+B[i];
}