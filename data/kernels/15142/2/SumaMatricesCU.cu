#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumaMatricesCU(int* A,int* B,int* C,int width){
int col=blockIdx.x*blockDim.x + threadIdx.x;//columnas
int row=blockIdx.y*blockDim.y + threadIdx.y;//filas

if((row<width)&&(col<width)){
C[row*width+col] = A[row*width+col]+B[row*width+col];
}
}