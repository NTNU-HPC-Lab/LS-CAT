#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// STD includes

// CUDA runtime

// Utilities and system includes


static // Print device properties
__global__ void readChannelKernel(unsigned char * image, unsigned char *channel, int imageW, int imageH, int channelToExtract, int numChannels) {
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;

int posIn = y  * (imageW*numChannels) + (x*numChannels) + channelToExtract;
int posOut = y * imageW + x;

channel[posOut] = image[posIn];


}