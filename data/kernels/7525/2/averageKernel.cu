#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// STD includes

// CUDA runtime

// Utilities and system includes


static // Print device properties
__global__ void averageKernel( unsigned char* inputChannel, unsigned char* outputChannel, int imageW, int imageH)
{
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numElements = ((2 * KERNEL_RADIUS) + 1) * ((2 * KERNEL_RADIUS) + 1);

unsigned int sum = 0;
for (int kY = -KERNEL_RADIUS; kY <= KERNEL_RADIUS; kY++) {
const int curY = y + kY;
if (curY < 0 || curY > imageH) {
continue;
}

for (int kX = -KERNEL_RADIUS; kX <= KERNEL_RADIUS; kX++) {
const int curX = x + kX;
if (curX < 0 || curX > imageW) {
continue;
}

const int curPosition = (curY * imageW + curX);
if (curPosition >= 0 && curPosition < (imageW * imageH)) {
sum += inputChannel[curPosition];
}
}
}
outputChannel[y * imageW + x] = (unsigned char)(sum / numElements);
}