#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// STD includes

// CUDA runtime

// Utilities and system includes


static // Print device properties
__global__ void writeChannelKernel( unsigned char* image, unsigned char* channel, int imageW, int imageH, int channelToMerge, int numChannels) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int posOut = y * (imageW*numChannels) + (x*numChannels) + channelToMerge;
int posIn = y * imageW + x;

image[posOut] = channel[posIn];

}