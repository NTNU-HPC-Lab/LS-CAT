#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "extract_snips2.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const double *Params = NULL;
hipMalloc(&Params, XSIZE*YSIZE);
const float *err = NULL;
hipMalloc(&err, XSIZE*YSIZE);
const int *st = NULL;
hipMalloc(&st, XSIZE*YSIZE);
const int *id = NULL;
hipMalloc(&id, XSIZE*YSIZE);
const int *counter = NULL;
hipMalloc(&counter, XSIZE*YSIZE);
const int *kk = NULL;
hipMalloc(&kk, XSIZE*YSIZE);
const int *iC = NULL;
hipMalloc(&iC, XSIZE*YSIZE);
const float *W = NULL;
hipMalloc(&W, XSIZE*YSIZE);
float *WU = NULL;
hipMalloc(&WU, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
extract_snips2<<<gridBlock,threadBlock>>>(Params,err,st,id,counter,kk,iC,W,WU);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
extract_snips2<<<gridBlock,threadBlock>>>(Params,err,st,id,counter,kk,iC,W,WU);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
extract_snips2<<<gridBlock,threadBlock>>>(Params,err,st,id,counter,kk,iC,W,WU);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}