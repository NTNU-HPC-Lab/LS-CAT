#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 5000, NrankMax = 6;
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *err, const int *ftype, float *x, int *st, int *id, int *counter){

int lockout, indx, tid, bid, NT, tid0,  j, t0;
volatile __shared__ float sdata[Nthreads+2*81+1];
bool flag=0;
float err0, Th;

lockout   = (int) Params[4] - 1;
tid 		= threadIdx.x;
bid 		= blockIdx.x;

NT      	=   (int) Params[0];
tid0 		= bid * blockDim.x ;
Th 		= (float) Params[2];

while(tid0<NT-Nthreads-lockout+1){
if (tid<2*lockout)
sdata[tid] = err[tid0 + tid];
if (tid0+tid+2*lockout<NT)
sdata[tid+2*lockout] = err[2*lockout + tid0 + tid];
else
sdata[tid+2*lockout] = 0.0f;

__syncthreads();

err0 = sdata[tid+lockout];
t0 = tid+lockout         + tid0;
if(err0 > Th*Th && t0<NT-lockout-1){
flag = 0;
for(j=-lockout;j<=lockout;j++)
if(sdata[tid+lockout+j]>err0){
flag = 1;
break;
}
if(flag==0){
indx = atomicAdd(&counter[0], 1);
if (indx<maxFR){
st[indx] = t0;
id[indx] = ftype[t0];
x[indx]  = err0;
}
}
}

tid0 = tid0 + blockDim.x * gridDim.x;
}
}