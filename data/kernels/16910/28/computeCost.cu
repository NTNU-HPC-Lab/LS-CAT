#include "hip/hip_runtime.h"
#include "includes.h"


//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////



//////////////////////////////////////////////////////////////////////////////////////////







//////////////////////////////////////////////////////////////////////////////////////////
__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, const bool *match, const int *iC, const int *call, float *cmax){

int NrankPC,j, NchanNear, tid, bid, Nspikes, Nthreads, k, my_chan, this_chan, Nchan;
float xsum = 0.0f, Ci, lam;

Nspikes               = (int) Params[0];
NrankPC             = (int) Params[1];
Nthreads              = blockDim.x;
lam                   = (float) Params[5];
NchanNear             = (int) Params[6];
Nchan                 = (int) Params[7];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

while(tid<Nspikes){
my_chan = call[tid];
if (match[my_chan + bid * Nchan]){
xsum = 0.0f;
for (k=0;k<NchanNear;k++)
for(j=0;j<NrankPC;j++){
this_chan = iC[k + my_chan * NchanNear];
xsum += uproj[j + NrankPC * k + NrankPC*NchanNear * tid] *
W[j + NrankPC * this_chan +  NrankPC*Nchan * bid];
}
Ci = max(0.0f, xsum) + lam/mu[bid];

cmax[tid + bid*Nspikes] = Ci * Ci / (1.0f + lam/(mu[bid] * mu[bid])) - lam;
}
tid+= Nthreads;
}
}