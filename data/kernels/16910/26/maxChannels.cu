#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max=81, NchanMax = 17;

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void maxChannels(const double *Params, const float *dataraw, const float *data, const int *iC, int *st, int *id, int *counter){

int nt0, indx, tid, tid0, i, bid, NT, Nchan, NchanNear,j,iChan, nt0min;
double Cf, d;
float spkTh;
bool flag;

NT 		= (int) Params[0];
Nchan     = (int) Params[1];
NchanNear = (int) Params[2];
nt0       = (int) Params[3];
nt0min    = (int) Params[4];
spkTh    = (float) Params[5];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

tid0 = tid + bid * blockDim.x;
while (tid0<NT-nt0-nt0min){
for (i=0; i<Nchan;i++){
iChan = iC[0 + NchanNear * i];
Cf    = (double) data[tid0 + NT * iChan];
flag = true;

for(j=1; j<NchanNear; j++){
iChan = iC[j+ NchanNear * i];
if (data[tid0 + NT * iChan] > Cf){
flag = false;
break;
}
}

if (flag){
iChan = iC[NchanNear * i];
if (Cf>spkTh){
d = (double) dataraw[tid0+nt0min-1 + NT*iChan]; //
if (d > Cf-1e-6){
// this is a hit, atomicAdd and return spikes
indx = atomicAdd(&counter[0], 1);
if (indx<maxFR){
st[indx] = tid0;
id[indx] = iChan;
}
}
}
}
}
tid0 += blockDim.x * gridDim.x;
}
}