#include "hip/hip_runtime.h"
#include "includes.h"


//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////



//////////////////////////////////////////////////////////////////////////////////////////







//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips_v3(const double *Params, const int *ioff, const int *id, const float *uproj, const float *cmax, float *bigArray){


// jic, version to work with Nfeatures threads
// have made a big array of Nfeature*NfeatW*Nfilters so projections
// onto each Nfeature can be summed without collisions
// after running this, need to sum up each set of Nfeature subArrays
// to calculate the final NfeatW*Nfilters array

int tid, bid, ind, Nspikes, Nfeatures, NfeatW;
float xsum = 0.0f;

Nspikes               = (int) Params[0];
Nfeatures             = (int) Params[1];
NfeatW                = (int) Params[4];

tid       = threadIdx.x;      //feature index
bid 		= blockIdx.x;       //filter index





for(ind=0; ind<Nspikes;ind++) {

if (id[ind]==bid){
//uproj is Nfeatures x Nspikes
xsum = uproj[tid + Nfeatures * ind];
//add this to the Nfeature-th array of NfeatW at the offset for this spike
bigArray[ioff[ind] + tid + tid*NfeatW + Nfeatures*NfeatW * bid] +=  xsum;
}  //end of if block for  match
}     //end of loop over spikes

}