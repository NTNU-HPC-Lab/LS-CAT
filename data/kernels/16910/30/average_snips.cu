#include "hip/hip_runtime.h"
#include "includes.h"


//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////



//////////////////////////////////////////////////////////////////////////////////////////







//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU){

int my_chan, this_chan, tidx, tidy, bid, ind, Nspikes, NrankPC, NchanNear, Nchan;
float xsum = 0.0f;

Nspikes               = (int) Params[0];
NrankPC             = (int) Params[1];
Nchan                = (int) Params[7];
NchanNear             = (int) Params[6];

tidx 		= threadIdx.x;
tidy 		= threadIdx.y;
bid 		= blockIdx.x;

for(ind=0; ind<Nspikes;ind++)
if (id[ind]==bid){
my_chan = call[ind];
this_chan = iC[tidy + NchanNear * my_chan];
xsum = uproj[tidx + NrankPC*tidy +  NrankPC*NchanNear * ind];
WU[tidx + NrankPC*this_chan + NrankPC*Nchan * bid] +=  xsum;
}
}