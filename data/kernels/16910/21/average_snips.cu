#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

// THIS UPDATE DOES NOT UPDATE ELOSS?
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////






//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const double *Params, const int *st, const int *id,  const float *x, const float *y,  const int *counter, const float *dataraw, const float *W, const float *U, double *WU, int *nsp, const float *mu, const float *z){

int nt0, tidx, tidy, bid, NT, Nchan,k, Nrank, Nfilt;
int currInd;
float Th;
double  X, xsum;

NT        = (int) Params[0];
Nfilt    	=   (int) Params[1];
nt0       = (int) Params[4];
Nrank     = (int) Params[6];
Nchan     = (int) Params[9];

tidx 		= threadIdx.x;
bid 		= blockIdx.x;

//Th = 10.f;
Th 		= (float) Params[15];

// we need wPCA projections in here, and then to decide based on total

// idx is the time sort order of the spikes; the original order is a function
// of when threads complete in mexGetSpikes. Compilation of the sums for WU, sig, and dnextbest
// in a fixed order makes the calculation deterministic.

for(currInd=0; currInd<counter[0];currInd++) {
// only do this if the spike is "GOOD"
if (x[currInd]>Th){
if (id[currInd]==bid){
if (tidx==0 &&  threadIdx.y==0)
nsp[bid]++;

tidy 		= threadIdx.y;
while (tidy<Nchan){
X = 0.0f;
for (k=0;k<Nrank;k++)
X += W[tidx + bid* nt0 + nt0*Nfilt*k] *
U[tidy + bid * Nchan + Nchan*Nfilt*k];

xsum = dataraw[st[currInd]+tidx + NT * tidy] + y[currInd] * X;

//WU[tidx+tidy*nt0 + nt0*Nchan * bid] *= p[bid];
WU[tidx+tidy*nt0 + nt0*Nchan * bid] += (double) xsum;

tidy+=blockDim.y;

}        //end of while loop over channels
}               //end of if block for id == bid
}
}                  //end of for loop over spike indicies
}                      //end of function