#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void reNormalize(const double *Params, const double *A, const double *B, double *W, double *U, double *mu){

int Nfilt, nt0, tid, bid, Nchan,k, Nrank, imax, t, ishift, tmax;
double x, xmax, xshift, sgnmax;

volatile __shared__ double sW[NrankMax*nt0max], sU[NchanMax*NrankMax], sS[NrankMax+1],
sWup[nt0max*10];

nt0       = (int) Params[4];
Nchan     = (int) Params[9];
Nfilt     = (int) Params[1];
Nrank     = (int) Params[6];
tmax = (int) Params[11];
bid 	  = blockIdx.x;

tid 		= threadIdx.x;
for(k=0;k<Nrank;k++)
sW[tid + k*nt0] = W[tid + bid*nt0 + k*Nfilt*nt0];

while (tid<Nchan*Nrank){
sU[tid] = U[tid%Nchan + bid*Nchan  + (tid/Nchan)*Nchan*Nfilt];
tid += blockDim.x;
}

__syncthreads();

tid 		= threadIdx.x;
if (tid<Nrank){
x = 0.0f;
for (k=0; k<Nchan; k++)
x += sU[k + tid*Nchan] * sU[k + tid*Nchan];
sS[tid] = sqrt(x);
}
// no need to sync here
if (tid==0){
x = 0.0000001f;
for (k=0;k<Nrank;k++)
x += sS[k] * sS[k];
sS[Nrank] = sqrt(x);
mu[bid] = sqrt(x);
}

__syncthreads();

// now re-normalize U
tid 		= threadIdx.x;

while (tid<Nchan*Nrank){
U[tid%Nchan + bid*Nchan  + (tid/Nchan)*Nchan*Nfilt] = sU[tid] / sS[Nrank];
tid += blockDim.x;
}

/////////////
__syncthreads();

// now align W
xmax = 0.0f;
imax = 0;
for(t=0;t<nt0;t++)
if (abs(sW[t]) > xmax){
xmax = abs(sW[t]);
imax = t;
}

tid 		= threadIdx.x;
// shift by imax - tmax
for (k=0;k<Nrank;k++){
ishift = tid + (imax-tmax);
ishift = (ishift%nt0 + nt0)%nt0;

xshift = sW[ishift + k*nt0];
W[tid + bid*nt0 + k*nt0*Nfilt] = xshift;
}

__syncthreads();
for (k=0;k<Nrank;k++){
sW[tid + k*nt0] = W[tid + bid*nt0 + k*nt0*Nfilt];
}

/////////////
__syncthreads();

// now align W. first compute 10x subsample peak
tid 		= threadIdx.x;
if (tid<10){
sWup[tid] = 0;
for (t=0;t<nt0;t++)
sWup[tid] += A[tid + t*10] * sW[t];
}
__syncthreads();

xmax = 0.0f;
imax = 0;
sgnmax = 1.0f;
for(t=0;t<10;t++)
if (abs(sWup[t]) > xmax){
xmax = abs(sWup[t]);
imax = t;
sgnmax = copysign(1.0f, sWup[t]);
}

// interpolate by imax
for (k=0;k<Nrank;k++){
xshift = 0.0f;
for (t=0;t<nt0;t++)
xshift += B[tid + t*nt0 +nt0*nt0*imax] * sW[t + k*nt0];

if (k==0)
xshift = -xshift * sgnmax;

W[tid + bid*nt0 + k*nt0*Nfilt] = xshift;
}

}