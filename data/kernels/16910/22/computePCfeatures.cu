#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

// THIS UPDATE DOES NOT UPDATE ELOSS?
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////






//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	computePCfeatures(const double *Params, const int *counter, const float *dataraw,  const int *st, const int *id, const float *x, const float *W, const float *U, const float *mu, const int *iW, const int *iC, const float *wPCA, float *featPC){

volatile __shared__ float  sPCA[81 * NrankMax], sW[81 * NrankMax], sU[NchanMax * NrankMax];
volatile __shared__ int iU[NchanMax];

int bid, nt0, t, tidx, tidy, k, NT, ind, Nchan, NchanU, Nfilt, Nrank;
float X = 0.0f, Y = 0.0f;

NT        = (int) Params[0];
nt0       = (int) Params[4];
Nchan     = (int) Params[9];
Nfilt    	= (int) Params[1];
Nrank     = (int) Params[6];
NchanU    = (int) Params[10];

tidx 		= threadIdx.x;
tidy 		= threadIdx.y;
bid       = blockIdx.x;

if (tidy==0)
iU[tidx] = iC[tidx + NchanU * iW[bid]];
__syncthreads();

sU[tidx + tidy*NchanU]= U[iU[tidx] + Nchan * bid + Nchan * Nfilt * tidy];

while (tidx<nt0){
sW[tidx + tidy*nt0]  = W[tidx + bid*nt0 + Nfilt * nt0 * tidy];
sPCA[tidx + tidy*nt0]  = wPCA[tidx + nt0 * tidy];
tidx += blockDim.x;
}

tidx 		= threadIdx.x;
__syncthreads();

//   first, compute wPCA projections of the filter
Y = 0.0f;
for (k =0; k<Nrank; k++){
X = 0.0f;
for (t=0;t<nt0;t++)
X += sW[t + k*nt0] * sPCA[t + tidy * nt0];
Y += X * sU[tidx + k*NchanU];
}

//now for each matching spike, compute the features
for(ind=0; ind<counter[0];ind++)
if (id[ind]==bid){
X = Y * x[ind]; // - mu[bid]);
for (t=0;t<nt0; t++)
X  += dataraw[st[ind] + t + NT * iU[tidx]] * sPCA[t + nt0*tidy];
featPC[tidx + tidy*NchanU + ind * NchanU*Nrank] = X;
}
}