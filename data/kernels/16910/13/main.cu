#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "spaceFilterUpdate.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const double *Params = NULL;
hipMalloc(&Params, XSIZE*YSIZE);
const float *data = NULL;
hipMalloc(&data, XSIZE*YSIZE);
const float *U = NULL;
hipMalloc(&U, XSIZE*YSIZE);
const bool *UtU = NULL;
hipMalloc(&UtU, XSIZE*YSIZE);
const int *iC = NULL;
hipMalloc(&iC, XSIZE*YSIZE);
const int *iW = NULL;
hipMalloc(&iW, XSIZE*YSIZE);
float *dprod = NULL;
hipMalloc(&dprod, XSIZE*YSIZE);
const int *st = NULL;
hipMalloc(&st, XSIZE*YSIZE);
const int *id = NULL;
hipMalloc(&id, XSIZE*YSIZE);
const int *counter = NULL;
hipMalloc(&counter, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
spaceFilterUpdate<<<gridBlock,threadBlock>>>(Params,data,U,UtU,iC,iW,dprod,st,id,counter);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
spaceFilterUpdate<<<gridBlock,threadBlock>>>(Params,data,U,UtU,iC,iW,dprod,st,id,counter);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
spaceFilterUpdate<<<gridBlock,threadBlock>>>(Params,data,U,UtU,iC,iW,dprod,st,id,counter);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}