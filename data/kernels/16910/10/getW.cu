#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getW(const double *Params, double *wtw, double *W){

int Nfilt, nt0, tid, bid, i, t, Nrank,k, tmax;
double x, x0, xmax;
volatile __shared__ double sW[nt0max*NrankMax], swtw[nt0max*nt0max], xN[1];

nt0       = (int) Params[4];
Nrank       = (int) Params[6];
Nfilt    	=   (int) Params[1];
tmax = (int) Params[11];

tid 		= threadIdx.x;
bid 		= blockIdx.x;

for (k=0;k<nt0;k++)
swtw[tid + k*nt0] = wtw[tid + k*nt0 + bid * nt0 * nt0];
for (k=0;k<Nrank;k++)
sW[tid + k*nt0] = W[tid + bid * nt0  + k * nt0*Nfilt];
__syncthreads();


// for each svd
for(k=0;k<Nrank;k++){
for (i=0;i<100;i++){
// compute projection of wtw
x = 0.0f;
for (t=0;t<nt0;t++)
x+= swtw[tid + t*nt0] * sW[t + k*nt0];

__syncthreads();
if (i<99){
sW[tid + k*nt0] = x;
__syncthreads();

if (tid==0){
x0 = 0.00001f;
for(t=0;t<nt0;t++)
x0+= sW[t + k*nt0] * sW[t + k*nt0];
xN[0] = sqrt(x0);
}
__syncthreads();

sW[tid + k*nt0] = x/xN[0];
__syncthreads();
}
}

// now subtract off this svd from wtw
for (t=0;t<nt0;t++)
swtw[tid + t*nt0] -= sW[t+k*nt0] * x;

__syncthreads();
}


xmax = sW[tmax];
__syncthreads();

sW[tid] = - sW[tid] * copysign(1.0, xmax);

// now write W back
for (k=0;k<Nrank;k++)
W[tid + bid * nt0  + k * nt0*Nfilt] = sW[tid + k*nt0];

}