#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max=81, NchanMax = 17;

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const double *Params, const float *data, const float *W, float *conv_sig){
volatile __shared__ float  sW[81*NrankMax], sdata[Nthreads+81];
float x, y;
int tid, tid0, bid, i, nid, Nrank, NT, nt0;

tid 		= threadIdx.x;
bid 		= blockIdx.x;

NT      	=   (int) Params[0];
nt0       = (int) Params[3];
Nrank     = (int) Params[6];

if(tid<nt0*Nrank)
sW[tid]= W[tid];
__syncthreads();

tid0 = 0;
while (tid0<NT-Nthreads-nt0+1){
if (tid<nt0)
sdata[tid] = data[tid0 + tid+ NT*bid];

sdata[tid + nt0] = data[tid0 + tid + nt0 + NT*bid];
__syncthreads();

x = 0.0f;
for(nid=0;nid<Nrank;nid++){
y = 0.0f;
#pragma unroll 4
for(i=0;i<nt0;i++)
y    += sW[i + nid*nt0] * sdata[i+tid];

x += y*y;
}
conv_sig[tid0  + tid + NT*bid]   = sqrt(x);

tid0+=Nthreads;
__syncthreads();
}
}