#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max=81, NchanMax = 17;

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  computeProjections(const double *Params, const float *dataraw, const int *iC, const int *st, const int *id, const float *W, float *feat){

float x;
int tidx, nt0min, tidy, my_chan, this_chan, tid, bid, nt0, NchanNear, j, t, NT, NrankPC;
volatile __shared__ float sW[nt0max*NrankMax], sD[nt0max*NchanMax];

NT 		= (int) Params[0];
NchanNear = (int) Params[2];
nt0       = (int) Params[3];
NrankPC  = (int) Params[6];
nt0min    = (int) Params[4];

tidx = threadIdx.x;
tidy = threadIdx.y;
bid = blockIdx.x;

// move wPCA to shared memory
while (tidx<nt0){
sW[tidx + tidy*nt0] = W[tidx + tidy*nt0];
tidx+=blockDim.x;
}
tidx = threadIdx.x;

tid = tidx + tidy*blockDim.x;
// move raw data to shared memory
while (tid<nt0){
my_chan = id[bid];
for (j=0;j<NchanNear;j++){
this_chan = iC[j + NchanNear*my_chan];
sD[tid + nt0*j] = dataraw[tid + st[bid]+nt0min-1 + NT * this_chan];
}
tid+=blockDim.x*blockDim.y;
}
__syncthreads();

x = 0.0f;
for (t=0;t<nt0;t++)
x += sD[t + nt0*tidx] * sW[t + nt0*tidy];

feat[tidy + tidx*NrankPC + NrankPC*NchanNear*bid] = x;

}