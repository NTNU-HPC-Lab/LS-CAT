#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMult(int *M, int *N, int *P, int width)
{
int tid, tx, ty;

tx = blockIdx.x*blockDim.x + threadIdx.x;
ty = blockIdx.y*blockDim.y + threadIdx.y;
tid = ty*width + tx;
int Pv = 0, Mv = 0, Nv = 0;

for(int i = 0; i < width; i++) {
Mv = M[ty*width+i];
Nv = N[i*width+tx];
Pv += Mv * Nv;
}

P[tid] = Pv;
}