#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaImageSmooth(unsigned char *In, unsigned char *Out, int width, int height, int fsize)
{
int row, col, destIndex;

col = blockIdx.x*blockDim.x + threadIdx.x;
row = blockIdx.y*blockDim.y + threadIdx.y;
destIndex = row*width + col;
int frow, fcol;
float tmp = 0.0;

if(col < fsize/2 || col > width-fsize/2 || row < fsize-2 || row > width-fsize/2) {
Out[destIndex] = 0;
} else {
for(frow = -fsize/2; frow <= fsize/2; frow++) {
for(fcol = -fsize/2; fcol <= fsize/2; fcol++) {
tmp += (float)In[(row+frow)*width+(col+fcol)];
}
}
tmp /= (fsize*fsize);	// average
Out[destIndex] = (unsigned char)tmp;
}
}