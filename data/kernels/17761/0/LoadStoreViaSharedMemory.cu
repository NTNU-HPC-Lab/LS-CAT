#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_SHAREDSIZE	2048


__global__ void LoadStoreViaSharedMemory(int *In, int *Out)
{
#if 1
int LoadStoreSize = MAX_SHAREDSIZE/blockDim.x;
int beginIndex = threadIdx.x * LoadStoreSize;
int endIndex = beginIndex + LoadStoreSize;

// °øÀ¯ ¸Þ¸ð¸® ÇÒ´ç
__shared__ int SharedMemory[MAX_SHAREDSIZE];
int i;

for(i = beginIndex; i < endIndex; i++)
SharedMemory[i] = In[i];

__syncthreads();

for(i = beginIndex; i < endIndex; i++)
Out[i] = SharedMemory[i];

__syncthreads();
#else
__shared__ int SharedMemory[MAX_SHAREDSIZE];

int idx = blockDim.x*blockIdx.x+threadIdx.x;
SharedMemory[idx] = In[idx];
Out[idx] = SharedMemory[idx];
#endif
}