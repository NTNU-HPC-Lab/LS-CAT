#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point2_gpu(int b, int n, int m, int nsample, const float *xyz1, const float *xyz2, const float *radii, int *idx, int *pts_cnt) {
int batch_index = blockIdx.x;
xyz1 += n*3*batch_index;
xyz2 += m*3*batch_index;
radii += m*batch_index;
idx += m*nsample*batch_index;  // m clusters, each having nsamples
pts_cnt += m*batch_index; // counting how many unique points selected in local region

int index = threadIdx.x;
int stride = blockDim.x;

for (int j=index;j<m;j+=stride) {  // index of cluster
int cnt = 0;
for (int k=0;k<n;++k) {  // index of point
if (cnt == nsample)
break; // only pick the FIRST nsample points in the ball
float x2=xyz2[j*3+0];
float y2=xyz2[j*3+1];
float z2=xyz2[j*3+2];
float x1=xyz1[k*3+0];
float y1=xyz1[k*3+1];
float z1=xyz1[k*3+2];
float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
if (d<radii[j]) {
if (cnt==0) { // set ALL indices to -1, s.t. we know which points are padded
for (int l=0;l<nsample;++l)
idx[j*nsample+l] = k;
}
idx[j*nsample+cnt] = k;
cnt+=1;
}
}
pts_cnt[j] = cnt;
}
}