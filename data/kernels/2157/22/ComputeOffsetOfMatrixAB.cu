#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
*(output + blockIdx.x * N + i) = K_A_B - row_sum[blockIdx.x] - col_sum[i];
}
}