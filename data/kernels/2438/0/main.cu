#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "DeformablePSROIPoolForwardKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int count = 1;
const double *bottom_data = NULL;
hipMalloc(&bottom_data, XSIZE*YSIZE);
const double spatial_scale = 1;
const int channels = 1;
const int height = 1;
const int width = 1;
const int pooled_height = 1;
const int pooled_width = 1;
const double *bottom_rois = NULL;
hipMalloc(&bottom_rois, XSIZE*YSIZE);
const double *bottom_trans = NULL;
hipMalloc(&bottom_trans, XSIZE*YSIZE);
const int no_trans = 1;
const double trans_std = 1;
const int sample_per_part = 1;
const int output_dim = 1;
const int group_size = 1;
const int part_size = 1;
const int num_classes = 1;
const int channels_each_class = 1;
double *top_data = NULL;
hipMalloc(&top_data, XSIZE*YSIZE);
double *top_count = NULL;
hipMalloc(&top_count, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
DeformablePSROIPoolForwardKernel<<<gridBlock,threadBlock>>>(count,bottom_data,spatial_scale,channels,height,width,pooled_height,pooled_width,bottom_rois,bottom_trans,no_trans,trans_std,sample_per_part,output_dim,group_size,part_size,num_classes,channels_each_class,top_data,top_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
DeformablePSROIPoolForwardKernel<<<gridBlock,threadBlock>>>(count,bottom_data,spatial_scale,channels,height,width,pooled_height,pooled_width,bottom_rois,bottom_trans,no_trans,trans_std,sample_per_part,output_dim,group_size,part_size,num_classes,channels_each_class,top_data,top_count);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
DeformablePSROIPoolForwardKernel<<<gridBlock,threadBlock>>>(count,bottom_data,spatial_scale,channels,height,width,pooled_height,pooled_width,bottom_rois,bottom_trans,no_trans,trans_std,sample_per_part,output_dim,group_size,part_size,num_classes,channels_each_class,top_data,top_count);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}