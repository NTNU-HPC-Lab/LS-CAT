#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void patchmatch_r_argmax_kernel( float *conv, float *target, float *match, int *correspondence, int c1, int h1, int w1, int h2, int w2 )
{
int id1 = blockIdx.x * blockDim.x + threadIdx.x;
int size1 = h1 * w1, size2 = h2 * w2;

if (id1 < size1) {
//int x1 = id1 % w1, y1 = id1 / w1;
double conv_max = -1e20;

for (int y2 = 0; y2 < h2; y2++) {
for (int x2 = 0; x2 < w2; x2++) {
int id2 = y2 * w2 + x2;

int id = id1 * size2 + id2;
float conv_result = conv[id];

if (conv_result > conv_max) {
conv_max = conv_result;
correspondence[id1 * 2 + 0] = x2;
correspondence[id1 * 2 + 1] = y2;
for (int c = 0; c < c1; c++) {
match[c * size1 + id1] = target[c * size2 + id2];
}
}
}
}

}

return ;
}