#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void hist_remap2_kernel( float *I, int nI, float *mI, float *histJ, float *cumJ, float *_minJ, float *_maxJ, int nbins, float *_sortI, int *_idxI, float *R, int c, int h, int w )
{
int _id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;

if (_id < c * size) {
// _id = dc * size + id
int id = _id % size, dc = _id / size;

float minJ  = _minJ[dc];
float maxJ  = _maxJ[dc];
float stepJ = (maxJ - minJ) / nbins;

int idxI = _idxI[_id] - 1;
if (mI[idxI] < EPS)
return ;
int offset = h * w - nI;

int cdf = id - offset;

int s = 0;
int e = nbins - 1;
int m = (s + e) / 2;
int binIdx = -1;

while (s <= e) {
// special handling for range boundary
float cdf_e = m == nbins - 1 ?
cumJ[dc * nbins + m] + 0.5f :
cumJ[dc * nbins + m];
float cdf_s = m == 0         ?
-0.5f :
cumJ[dc * nbins + m - 1];

if (cdf >= cdf_e) {
s = m + 1;
m = (s + e) / 2;
} else if (cdf < cdf_s) {
e = m - 1;
m = (s + e) / 2;
} else {
binIdx = m;    break;
}
}

float hist  = histJ[dc * nbins + binIdx];
float cdf_e = cumJ[dc * nbins + binIdx];
float cdf_s = cdf_e - hist;
float ratio = MIN(MAX((cdf - cdf_s) / (hist + 1e-8), 0.0f), 1.0f);
float activation = minJ + (static_cast<float>(binIdx) + ratio) * stepJ;
R[dc * size + idxI] = activation;
}

return ;
}