#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
const int igpt =
(gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

if (igpt < n) {
dout[igpt] = zin[2 * igpt];
}
}