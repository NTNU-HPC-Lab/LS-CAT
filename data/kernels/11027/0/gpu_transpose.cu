#include "hip/hip_runtime.h"
#include "includes.h"
int row = 0;
int col = 0;
using namespace std;

__global__
__global__ void gpu_transpose(float *dst, float *A, int col, int row) {
int idx = threadIdx.x + blockIdx.x*blockDim.x;

if(idx<col){
for (int j=0; j<row; j++){
dst[j*col+idx] = A[idx*row+j];
}
}
}