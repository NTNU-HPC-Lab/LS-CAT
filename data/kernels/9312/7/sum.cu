#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a,int *b,int n)
{
int block=256*blockIdx.x;
int sum=0;
for(int i=block;i<min(block+256,n);i++)
{
sum=sum+a[i];
}
b[blockIdx.x]=sum;
}