#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;



__global__ void variance(int* n, double *x, double *mean)
{
int index = threadIdx.x;
int stride = blockDim.x;

for (int i = index; i < *n; i+= stride) {
x[i] = (x[i] - *mean) ;
x[i] = x[i] * x[i];
}
}