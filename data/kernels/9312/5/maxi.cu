#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxi(int * a, int * b, int n) {

int block = 256 * blockIdx.x;
int max = 0;

for (int i = block; i < min(256 + block, n); i++) {
if (max < a[i]) {
max = a[i];
}
}
b[blockIdx.x] = max;
}