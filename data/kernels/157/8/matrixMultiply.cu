#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
//@@ Insert code to implement matrix multiplication here
// Calculate the row index
int numRows = blockIdx.y*blockDim.y+threadIdx.y;
// Calculate the column index
int numColumns = blockIdx.x*blockDim.x+threadIdx.x;
if ((numRows < numARows) && (numColumns < numBColumns)) {
float Cval = 0.0;
// Each thread computes one element of the block sub-matrix
for (int k = 0; k < numBRows; ++k) {
Cval += A[numRows*numBRows+k]*B[numColumns+k*numBColumns];
}
C[numRows*numBColumns+numColumns] = Cval;
}
}