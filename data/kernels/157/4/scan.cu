#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float * input, float * output, int len) {
//@@ Modify the body of this function to complete the functionality of
//@@ the scan on the device
//@@ You may need multiple kernel calls; write your kernels before this
//@@ function and call them from here
__shared__ float scan_array[BLOCK_SIZE];
int global_id = threadIdx.x + blockDim.x * blockIdx.x;
if (global_id < len)
scan_array[threadIdx.x] = input[global_id];
else
scan_array[threadIdx.x] = 0;
__syncthreads();
int stride = 1;
while (stride < BLOCK_SIZE) {
int index = (threadIdx.x + 1) * stride * 2 - 1;
if (index < BLOCK_SIZE)
scan_array[index] += scan_array[index - stride];
stride = stride << 1;
__syncthreads();
}

for(int stride = BLOCK_SIZE >> 1; stride > 0; stride = stride >> 1) {
__syncthreads();
int index = (threadIdx.x + 1) * stride * 2 - 1;
if (index + stride < BLOCK_SIZE)
scan_array[index + stride] += scan_array[index];
}
__syncthreads();
if (global_id < len)
output[global_id] = scan_array[threadIdx.x];


if (global_id < BLOCK_SIZE) {
__syncthreads();
for (int block_idx = 1; block_idx <= (len / BLOCK_SIZE) ; ++block_idx) {
float offset = output[block_idx * BLOCK_SIZE - 1];
if ((threadIdx.x + block_idx * blockDim.x) < len)
output[threadIdx.x + block_idx * blockDim.x] += offset;
__syncthreads();
}
}
}