#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256












__global__ void hist_eq(unsigned char * deviceCharImg, float * output, float* cdf, float cdfmin, int size)
{
int bx = blockIdx.x;
int tx = threadIdx.x;


int i = tx+blockDim.x*bx;

if(i < size)
{
deviceCharImg[i] = min(max(255*(cdf[deviceCharImg[i]] - cdfmin)/(1 - cdfmin),0.0),255.0);

output[i] = (float) (deviceCharImg[i]/255.0);

}
}