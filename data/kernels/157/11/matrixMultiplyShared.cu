#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
//@@ Insert code to implement matrix multiplication here
//@@ You have to use shared memory for this MP
__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
int tx = threadIdx.x;
int ty = threadIdx.y;
int m = numARows;
int n = numBRows;
int k = numBColumns;

int numRows = blockIdx.y * blockDim.y + ty;
int numColumns = blockIdx.x * blockDim.x + tx;
float Cval = 0.0;

//Loading A and B elements and doing Boundary Check
for(int t = 0; t < (n-1)/TILE_WIDTH + 1; t++) {

if((numRows < numARows) && (t*TILE_WIDTH+tx < n)) {
ds_A[ty][tx] = A[numRows*n + t*TILE_WIDTH+tx];
} else {
ds_A[ty][tx] = 0.0;
}

if((numColumns < k) && (t*TILE_WIDTH+ty < n)) {
ds_B[ty][tx] = B[(t*TILE_WIDTH+ty)*k + numColumns];
} else {
ds_B[ty][tx] = 0.0;
}
__syncthreads();

for(int i = 0; i < TILE_WIDTH; i++) {
Cval += ds_A[ty][i] * ds_B[i][tx];
}
__syncthreads();
}

if(numRows < m && numColumns < k) {
C[numRows*k + numColumns] = Cval;
}
}