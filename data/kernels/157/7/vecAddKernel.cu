#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if(i<n) //boundary check for threads.
{
C[i] = A[i] + B[i];
}
}