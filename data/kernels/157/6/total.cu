#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void total(float * input, float * output, int len) {
//@@ Load a segment of the input vector into shared memory
__shared__ float partialSum[2 * BLOCK_SIZE];
unsigned int tx = threadIdx.x;
unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

if ((start + tx) < len) {
partialSum[tx] = input[start + tx];
}
else {
partialSum[tx] = 0.0;
}
if ((start + BLOCK_SIZE + tx) < len) {
partialSum[BLOCK_SIZE + tx] = input[start + BLOCK_SIZE + tx];
}
else {
partialSum[BLOCK_SIZE + tx] = 0.0;
}

//@@ Traverse the reduction tree
for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride/=2) {
__syncthreads();
if (tx < stride) {
partialSum[tx] += partialSum[tx + stride];
}
}

//@@ Write the computed sum of the block to the output vector at the
//@@ correct index
// Boundary condition is handled by filling “identity value (0 for sum)”
// into the shared memory of the last block
if (tx == 0) {
output[blockIdx.x] = partialSum[0];
}

}