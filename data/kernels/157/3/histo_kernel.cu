#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256












__global__ void histo_kernel(unsigned char * buffer, unsigned int * histo, long size)
{
//  compute histogram with a private version in each block
__shared__ unsigned int histo_private[HISTOGRAM_LENGTH];

int bx = blockIdx.x;
int tx = threadIdx.x;

//  index of current pixel
int index = tx+bx*blockDim.x;

//  set initial values of histogram to zero
if (tx < HISTOGRAM_LENGTH) histo_private[tx] = 0;

__syncthreads();


int stride = blockDim.x*gridDim.x;

//iterate to add values
while (index < stride)
{
atomicAdd(&(histo_private[buffer[index]]), 1);
index += stride;
}

__syncthreads();

//copy private histogram to device histogram
if(tx<256)
{
atomicAdd(&(histo[tx]), histo_private[tx]);
}
}