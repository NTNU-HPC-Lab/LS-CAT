#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
//@@ Insert code to implement vector addition here
int idx  = threadIdx.x + blockDim.x * blockIdx.x;
if (idx  < len) {
out[idx ] = in1[idx] + in2[idx];
}
}