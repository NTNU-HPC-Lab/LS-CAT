#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256












__global__ void convertToGrayScale(unsigned char * ucharImg, unsigned char * grayImg, int width, int height)
{

int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

int row = by*blockDim.y+ty;
int col = bx*blockDim.x+tx;
int index = row*width + col;

if(row < height && col < width)
{
grayImg[index] = (unsigned char) (0.21*ucharImg[index*3] + 0.71*ucharImg[index*3 + 1] + 0.07*ucharImg[index*3 + 2]);
}

}