#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256












__global__ void convertToChar(float * input, unsigned char * ucharInput, int width, int height)
{
int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

int row = by*blockDim.y+ty;
int col = bx*blockDim.x+tx;
int index = row*width + col;

if(row < height && col < width)
{
ucharInput[index*3]   = (unsigned char) (255 * input[index*3]); //r
ucharInput[index*3+1] = (unsigned char) (255 * input[index*3+1]); //g
ucharInput[index*3+2] = (unsigned char) (255 * input[index*3+2]); //b
}


}