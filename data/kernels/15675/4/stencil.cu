#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil(int *in, int *out)
{
int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
int value = 0;
for(int offset = -RADIUS; offset <= RADIUS; offset++)
value += in[globIdx + offset];
out[globIdx] = value;
}