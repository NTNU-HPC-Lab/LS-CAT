#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vevAdd(int N, float *a, float *b, float *c)
{
// work idex, 在launch kernel的时候指定维度
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < N)
{
c[idx] = a[idx] + b[idx];
}
}