#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPUmemo( float *data, int pts )
{
__shared__ float* trace;

trace = (float *)malloc(pts*sizeof(float));
int Blocks;
for( Blocks = 0; Blocks < gridDim.x; Blocks++ )
{
trace[threadIdx.x] = data[threadIdx.x + Blocks*pts];
}
}