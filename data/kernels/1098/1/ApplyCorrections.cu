#include "hip/hip_runtime.h"
#include "includes.h"
////////////////////////////////////////////////////////////////////////////////
/*
Hologram generating algorithms for CUDA Devices

Copyright 2009, 2010, 2011, 2012 Martin Persson
martin.persson@physics.gu.se

This file is part of GenerateHologramCUDA.

GenerateHologramCUDA is free software: you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public License as published
by the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

GenerateHologramCUDA is distributed in the hope that it will be
useful, but WITHOUT ANY WARRANTY; without even the implied warranty
of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains two different algorithms for
//hologram generation. The last parameter in the function call selects which
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//2: Weighted Gerchberg-Saxton algorithm using Fast Fourier Transforms (2D)
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster.
//     (0) is automatically selected if the number of spots is < 3.
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007)
//
//The original algorithm has been modified to allow variable spot amplitudes
////////////////////////////////////////////////////////////////////////////////
//Naming convention for variables:
//-The prefix indicates where data is located
//--In host functions:		h = host memory
//				d = device memory
//				c = constant memory
//--In global functions:	g = global memory
//				s = shared memory
//				c = constant memory
//				no prefix = registers
//-The suffix indicates the data type, no suffix usually indicates an iteger
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Improve convergence of the GS algorithms for 2 spots.							*done
//-Compensate spot intensities for distance from center of field.					*done
//-Put all arguments for device functions and trap positions in constant memory.	*done
// (Requires all functions to be moved into the same file or the use of some
// workaround found on nVidia forum)
//-Put pSLMstart and aLaser in texture memory (may not improve performance on Fermi devices)
//-Use "zero-copy" to transfer pSLM to host.
//-Rename functions and variables for consistency and readability
//-Allow variable spot phases for Lenses and Prisms
////////////////////////////////////////////////////////////////////////////////

//#define M_CUDA_DEBUG			   //activates a number of custom debug macros//
float dt_milliseconds;
hipEvent_t start, stop;
////////////////////////////////////////////////////////////////////////////////
//Includes
////////////////////////////////////////////////////////////////////////////////

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 1024	//decrease this if your GPU keeps running out of memory
#define BLOCK_SIZE 256	//should be a power of 2
#define SLM_SIZE 512
#if ((SLM_SIZE==16)||(SLM_SIZE==32)||(SLM_SIZE==64)||(SLM_SIZE==128)||(SLM_SIZE==256)||(SLM_SIZE==512)||(SLM_SIZE==1024)||(SLM_SIZE==2048))
#define SLMPOW2			//Uses bitwise modulu operations if the SLM size is a power of 2
#endif

////////////////////////////////////////////////////////////////////////////////
// forward declarations
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//Set correction parameters
////////////////////////////////////////////////////////////////////////////////
__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c)
{
switch (c_N_PolLUTCoeff[0])	{
case 120:
return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*X*X + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*Y*Y + s_c[8]*Y*phase2pi + s_c[9]*phase2pi*phase2pi + s_c[10]*X*X*X + s_c[11]*X*X*Y + s_c[12]*X*X*phase2pi + s_c[13]*X*Y*Y + s_c[14]*X*Y*phase2pi + s_c[15]*X*phase2pi*phase2pi + s_c[16]*Y*Y*Y + s_c[17]*Y*Y*phase2pi + s_c[18]*Y*phase2pi*phase2pi + s_c[19]*phase2pi*phase2pi*phase2pi + s_c[20]*X*X*X*X + s_c[21]*X*X*X*Y + s_c[22]*X*X*X*phase2pi + s_c[23]*X*X*Y*Y + s_c[24]*X*X*Y*phase2pi + s_c[25]*X*X*phase2pi*phase2pi + s_c[26]*X*Y*Y*Y + s_c[27]*X*Y*Y*phase2pi + s_c[28]*X*Y*phase2pi*phase2pi + s_c[29]*X*phase2pi*phase2pi*phase2pi + s_c[30]*Y*Y*Y*Y + s_c[31]*Y*Y*Y*phase2pi + s_c[32]*Y*Y*phase2pi*phase2pi + s_c[33]*Y*phase2pi*phase2pi*phase2pi + s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi + s_c[35]*X*X*X*X*X + s_c[36]*X*X*X*X*Y + s_c[37]*X*X*X*X*phase2pi + s_c[38]*X*X*X*Y*Y + s_c[39]*X*X*X*Y*phase2pi + s_c[40]*X*X*X*phase2pi*phase2pi + s_c[41]*X*X*Y*Y*Y + s_c[42]*X*X*Y*Y*phase2pi + s_c[43]*X*X*Y*phase2pi*phase2pi + s_c[44]*X*X*phase2pi*phase2pi*phase2pi + s_c[45]*X*Y*Y*Y*Y + s_c[46]*X*Y*Y*Y*phase2pi + s_c[47]*X*Y*Y*phase2pi*phase2pi + s_c[48]*X*Y*phase2pi*phase2pi*phase2pi + s_c[49]*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[50]*Y*Y*Y*Y*Y + s_c[51]*Y*Y*Y*Y*phase2pi + s_c[52]*Y*Y*Y*phase2pi*phase2pi + s_c[53]*Y*Y*phase2pi*phase2pi*phase2pi + s_c[54]*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[55]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[56]*X*X*X*X*X*X + s_c[57]*X*X*X*X*X*Y + s_c[58]*X*X*X*X*X*phase2pi + s_c[59]*X*X*X*X*Y*Y + s_c[60]*X*X*X*X*Y*phase2pi + s_c[61]*X*X*X*X*phase2pi*phase2pi + s_c[62]*X*X*X*Y*Y*Y + s_c[63]*X*X*X*Y*Y*phase2pi + s_c[64]*X*X*X*Y*phase2pi*phase2pi + s_c[65]*X*X*X*phase2pi*phase2pi*phase2pi + s_c[66]*X*X*Y*Y*Y*Y + s_c[67]*X*X*Y*Y*Y*phase2pi + s_c[68]*X*X*Y*Y*phase2pi*phase2pi + s_c[69]*X*X*Y*phase2pi*phase2pi*phase2pi + s_c[70]*X*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[71]*X*Y*Y*Y*Y*Y + s_c[72]*X*Y*Y*Y*Y*phase2pi + s_c[73]*X*Y*Y*Y*phase2pi*phase2pi + s_c[74]*X*Y*Y*phase2pi*phase2pi*phase2pi + s_c[75]*X*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[76]*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[77]*Y*Y*Y*Y*Y*Y + s_c[78]*Y*Y*Y*Y*Y*phase2pi + s_c[79]*Y*Y*Y*Y*phase2pi*phase2pi + s_c[80]*Y*Y*Y*phase2pi*phase2pi*phase2pi + s_c[81]*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[82]*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[83]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[84]*X*X*X*X*X*X*X + s_c[85]*X*X*X*X*X*X*Y + s_c[86]*X*X*X*X*X*X*phase2pi + s_c[87]*X*X*X*X*X*Y*Y + s_c[88]*X*X*X*X*X*Y*phase2pi + s_c[89]*X*X*X*X*X*phase2pi*phase2pi + s_c[90]*X*X*X*X*Y*Y*Y + s_c[91]*X*X*X*X*Y*Y*phase2pi + s_c[92]*X*X*X*X*Y*phase2pi*phase2pi + s_c[93]*X*X*X*X*phase2pi*phase2pi*phase2pi + s_c[94]*X*X*X*Y*Y*Y*Y + s_c[95]*X*X*X*Y*Y*Y*phase2pi + s_c[96]*X*X*X*Y*Y*phase2pi*phase2pi + s_c[97]*X*X*X*Y*phase2pi*phase2pi*phase2pi + s_c[98]*X*X*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[99]*X*X*Y*Y*Y*Y*Y + s_c[100]*X*X*Y*Y*Y*Y*phase2pi + s_c[101]*X*X*Y*Y*Y*phase2pi*phase2pi + s_c[102]*X*X*Y*Y*phase2pi*phase2pi*phase2pi + s_c[103]*X*X*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[104]*X*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[105]*X*Y*Y*Y*Y*Y*Y + s_c[106]*X*Y*Y*Y*Y*Y*phase2pi + s_c[107]*X*Y*Y*Y*Y*phase2pi*phase2pi + s_c[108]*X*Y*Y*Y*phase2pi*phase2pi*phase2pi + s_c[109]*X*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[110]*X*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[111]*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[112]*Y*Y*Y*Y*Y*Y*Y + s_c[113]*Y*Y*Y*Y*Y*Y*phase2pi + s_c[114]*Y*Y*Y*Y*Y*phase2pi*phase2pi + s_c[115]*Y*Y*Y*Y*phase2pi*phase2pi*phase2pi + s_c[116]*Y*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[117]*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[118]*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[119]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi);
case 84:
return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*X*X + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*Y*Y + s_c[8]*Y*phase2pi + s_c[9]*phase2pi*phase2pi + s_c[10]*X*X*X + s_c[11]*X*X*Y + s_c[12]*X*X*phase2pi + s_c[13]*X*Y*Y + s_c[14]*X*Y*phase2pi + s_c[15]*X*phase2pi*phase2pi + s_c[16]*Y*Y*Y + s_c[17]*Y*Y*phase2pi + s_c[18]*Y*phase2pi*phase2pi + s_c[19]*phase2pi*phase2pi*phase2pi + s_c[20]*X*X*X*X + s_c[21]*X*X*X*Y + s_c[22]*X*X*X*phase2pi + s_c[23]*X*X*Y*Y + s_c[24]*X*X*Y*phase2pi + s_c[25]*X*X*phase2pi*phase2pi + s_c[26]*X*Y*Y*Y + s_c[27]*X*Y*Y*phase2pi + s_c[28]*X*Y*phase2pi*phase2pi + s_c[29]*X*phase2pi*phase2pi*phase2pi + s_c[30]*Y*Y*Y*Y + s_c[31]*Y*Y*Y*phase2pi + s_c[32]*Y*Y*phase2pi*phase2pi + s_c[33]*Y*phase2pi*phase2pi*phase2pi + s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi + s_c[35]*X*X*X*X*X + s_c[36]*X*X*X*X*Y + s_c[37]*X*X*X*X*phase2pi + s_c[38]*X*X*X*Y*Y + s_c[39]*X*X*X*Y*phase2pi + s_c[40]*X*X*X*phase2pi*phase2pi + s_c[41]*X*X*Y*Y*Y + s_c[42]*X*X*Y*Y*phase2pi + s_c[43]*X*X*Y*phase2pi*phase2pi + s_c[44]*X*X*phase2pi*phase2pi*phase2pi + s_c[45]*X*Y*Y*Y*Y + s_c[46]*X*Y*Y*Y*phase2pi + s_c[47]*X*Y*Y*phase2pi*phase2pi + s_c[48]*X*Y*phase2pi*phase2pi*phase2pi + s_c[49]*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[50]*Y*Y*Y*Y*Y + s_c[51]*Y*Y*Y*Y*phase2pi + s_c[52]*Y*Y*Y*phase2pi*phase2pi + s_c[53]*Y*Y*phase2pi*phase2pi*phase2pi + s_c[54]*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[55]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[56]*X*X*X*X*X*X + s_c[57]*X*X*X*X*X*Y + s_c[58]*X*X*X*X*X*phase2pi + s_c[59]*X*X*X*X*Y*Y + s_c[60]*X*X*X*X*Y*phase2pi + s_c[61]*X*X*X*X*phase2pi*phase2pi + s_c[62]*X*X*X*Y*Y*Y + s_c[63]*X*X*X*Y*Y*phase2pi + s_c[64]*X*X*X*Y*phase2pi*phase2pi + s_c[65]*X*X*X*phase2pi*phase2pi*phase2pi + s_c[66]*X*X*Y*Y*Y*Y + s_c[67]*X*X*Y*Y*Y*phase2pi + s_c[68]*X*X*Y*Y*phase2pi*phase2pi + s_c[69]*X*X*Y*phase2pi*phase2pi*phase2pi + s_c[70]*X*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[71]*X*Y*Y*Y*Y*Y + s_c[72]*X*Y*Y*Y*Y*phase2pi + s_c[73]*X*Y*Y*Y*phase2pi*phase2pi + s_c[74]*X*Y*Y*phase2pi*phase2pi*phase2pi + s_c[75]*X*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[76]*X*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[77]*Y*Y*Y*Y*Y*Y + s_c[78]*Y*Y*Y*Y*Y*phase2pi + s_c[79]*Y*Y*Y*Y*phase2pi*phase2pi + s_c[80]*Y*Y*Y*phase2pi*phase2pi*phase2pi + s_c[81]*Y*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[82]*Y*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi + s_c[83]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi);
case 56:
return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*X*X + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*Y*Y + s_c[8]*Y*phase2pi + s_c[9]*phase2pi*phase2pi + s_c[10]*X*X*X + s_c[11]*X*X*Y + s_c[12]*X*X*phase2pi + s_c[13]*X*Y*Y + s_c[14]*X*Y*phase2pi + s_c[15]*X*phase2pi*phase2pi + s_c[16]*Y*Y*Y + s_c[17]*Y*Y*phase2pi + s_c[18]*Y*phase2pi*phase2pi + s_c[19]*phase2pi*phase2pi*phase2pi + s_c[20]*X*X*X*X + s_c[21]*X*X*X*Y + s_c[22]*X*X*X*phase2pi + s_c[23]*X*X*Y*Y + s_c[24]*X*X*Y*phase2pi + s_c[25]*X*X*phase2pi*phase2pi + s_c[26]*X*Y*Y*Y + s_c[27]*X*Y*Y*phase2pi + s_c[28]*X*Y*phase2pi*phase2pi + s_c[29]*X*phase2pi*phase2pi*phase2pi + s_c[30]*Y*Y*Y*Y + s_c[31]*Y*Y*Y*phase2pi + s_c[32]*Y*Y*phase2pi*phase2pi + s_c[33]*Y*phase2pi*phase2pi*phase2pi + s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi + s_c[35]*X*X*X*X*X + s_c[36]*X*X*X*X*Y + s_c[37]*X*X*X*X*phase2pi + s_c[38]*X*X*X*Y*Y + s_c[39]*X*X*X*Y*phase2pi + s_c[40]*X*X*X*phase2pi*phase2pi + s_c[41]*X*X*Y*Y*Y + s_c[42]*X*X*Y*Y*phase2pi + s_c[43]*X*X*Y*phase2pi*phase2pi + s_c[44]*X*X*phase2pi*phase2pi*phase2pi + s_c[45]*X*Y*Y*Y*Y + s_c[46]*X*Y*Y*Y*phase2pi + s_c[47]*X*Y*Y*phase2pi*phase2pi + s_c[48]*X*Y*phase2pi*phase2pi*phase2pi + s_c[49]*X*phase2pi*phase2pi*phase2pi*phase2pi + s_c[50]*Y*Y*Y*Y*Y + s_c[51]*Y*Y*Y*Y*phase2pi + s_c[52]*Y*Y*Y*phase2pi*phase2pi + s_c[53]*Y*Y*phase2pi*phase2pi*phase2pi + s_c[54]*Y*phase2pi*phase2pi*phase2pi*phase2pi + s_c[55]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi);
case 35:
return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*X*X + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*Y*Y + s_c[8]*Y*phase2pi + s_c[9]*phase2pi*phase2pi + s_c[10]*X*X*X + s_c[11]*X*X*Y + s_c[12]*X*X*phase2pi + s_c[13]*X*Y*Y + s_c[14]*X*Y*phase2pi + s_c[15]*X*phase2pi*phase2pi + s_c[16]*Y*Y*Y + s_c[17]*Y*Y*phase2pi + s_c[18]*Y*phase2pi*phase2pi + s_c[19]*phase2pi*phase2pi*phase2pi + s_c[20]*X*X*X*X + s_c[21]*X*X*X*Y + s_c[22]*X*X*X*phase2pi + s_c[23]*X*X*Y*Y + s_c[24]*X*X*Y*phase2pi + s_c[25]*X*X*phase2pi*phase2pi + s_c[26]*X*Y*Y*Y + s_c[27]*X*Y*Y*phase2pi + s_c[28]*X*Y*phase2pi*phase2pi + s_c[29]*X*phase2pi*phase2pi*phase2pi + s_c[30]*Y*Y*Y*Y + s_c[31]*Y*Y*Y*phase2pi + s_c[32]*Y*Y*phase2pi*phase2pi + s_c[33]*Y*phase2pi*phase2pi*phase2pi + s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi);
case 20:
return (unsigned char)(s_c[0] + s_c[1]*X + s_c[2]*Y + s_c[3]*phase2pi + s_c[4]*X*X + s_c[5]*X*Y + s_c[6]*X*phase2pi + s_c[7]*Y*Y + s_c[8]*Y*phase2pi + s_c[9]*phase2pi*phase2pi + s_c[10]*X*X*X + s_c[11]*X*X*Y + s_c[12]*X*X*phase2pi + s_c[13]*X*Y*Y + s_c[14]*X*Y*phase2pi + s_c[15]*X*phase2pi*phase2pi + s_c[16]*Y*Y*Y + s_c[17]*Y*Y*phase2pi + s_c[18]*Y*phase2pi*phase2pi + s_c[19]*phase2pi*phase2pi*phase2pi);
default:
return 0;
}
}
__device__ int getYint(int index, int X_int)
{
#ifdef SLMPOW2
int Y_int = (index-X_int)>>c_log2data_w[0];
#else
int Y_int = (float)(floor((float)index/c_data_w_f[0]));
#endif
return Y_int;
}
__device__ int getXint(int index)
{
#ifdef SLMPOW2
int X_int = index&(c_data_w[0]-1);
#else
float X_int= index%c_data_w[0];
#endif
return X_int;
}
__device__ float ApplyAberrationCorrection(float pSpot, float correction)
{
pSpot = pSpot - correction;		//apply correction
return (pSpot - (2.0f*M_PI) * floor((pSpot+M_PI) / (2.0f*M_PI))); //apply mod([-pi, pi], pSpot)
}
__device__ int phase2int32(float phase2pi)
{
return (int)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ unsigned char phase2uc(float phase2pi)
{
return (unsigned char)floor((phase2pi + M_PI)*256.0f / (2.0f * M_PI));
}
__device__ float uc2phase(float uc)
{
return (float)uc*2.0f*M_PI/256.0f - M_PI;
}
__global__ void ApplyCorrections(unsigned char *g_pSLM_uc, unsigned char *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
int tid = threadIdx.x;
int idx = blockIdx.x * blockDim.x + threadIdx.x;
float pSLM2pi_f = uc2phase(g_pSLM_uc[idx]);
if (c_useAberrationCorr_b[0])
pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);

if (c_usePolLUT_b[0])
{
int X_int = getXint(idx);
int Y_int = getYint(idx, X_int);
float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);
__shared__ float s_LUTcoeff[120];
if (tid < c_N_PolLUTCoeff[0])
s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
__syncthreads();
g_pSLM_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff);
}
else if (c_applyLUT_b[0])
{
__shared__ unsigned char s_LUT[256];
if (tid < 256)
s_LUT[tid] = g_LUT[tid];
__syncthreads();
g_pSLM_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
}
else
g_pSLM_uc[idx] = phase2uc(pSLM2pi_f);
}