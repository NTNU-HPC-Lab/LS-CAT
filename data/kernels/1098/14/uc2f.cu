#include "hip/hip_runtime.h"
#include "includes.h"
////////////////////////////////////////////////////////////////////////////////
/*
Hologram generating algorithms for CUDA Devices

Copyright 2009, 2010, 2011, 2012 Martin Persson
martin.persson@physics.gu.se

This file is part of GenerateHologramCUDA.

GenerateHologramCUDA is free software: you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public License as published
by the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

GenerateHologramCUDA is distributed in the hope that it will be
useful, but WITHOUT ANY WARRANTY; without even the implied warranty
of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.
*/
///////////////////////////////////////////////////////////////////////////////////
//The function "GenerateHologram" contains two different algorithms for
//hologram generation. The last parameter in the function call selects which
//one to use:
//0: Complex addition of "Lenses and Prisms", no optimization (3D)
//1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
//2: Weighted Gerchberg-Saxton algorithm using Fast Fourier Transforms (2D)
//-(0) produces optimal holograms for 1 or 2 traps and is significantly faster.
//     (0) is automatically selected if the number of spots is < 3.
////////////////////////////////////////////////////////////////////////////////
//Fresnel propagation based algorithm (1) described in:
//Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
//"Computer generation of optimal holograms for optical trap arrays"
//Opt. Express 15, 1913-1922 (2007)
//
//The original algorithm has been modified to allow variable spot amplitudes
////////////////////////////////////////////////////////////////////////////////
//Naming convention for variables:
//-The prefix indicates where data is located
//--In host functions:		h = host memory
//				d = device memory
//				c = constant memory
//--In global functions:	g = global memory
//				s = shared memory
//				c = constant memory
//				no prefix = registers
//-The suffix indicates the data type, no suffix usually indicates an iteger
////////////////////////////////////////////////////////////////////////////////
//Possible improvements:
//-Improve convergence of the GS algorithms for 2 spots.							*done
//-Compensate spot intensities for distance from center of field.					*done
//-Put all arguments for device functions and trap positions in constant memory.	*done
// (Requires all functions to be moved into the same file or the use of some
// workaround found on nVidia forum)
//-Put pSLMstart and aLaser in texture memory (may not improve performance on Fermi devices)
//-Use "zero-copy" to transfer pSLM to host.
//-Rename functions and variables for consistency and readability
//-Allow variable spot phases for Lenses and Prisms
////////////////////////////////////////////////////////////////////////////////

//#define M_CUDA_DEBUG			   //activates a number of custom debug macros//
float dt_milliseconds;
hipEvent_t start, stop;
////////////////////////////////////////////////////////////////////////////////
//Includes
////////////////////////////////////////////////////////////////////////////////

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 1024	//decrease this if your GPU keeps running out of memory
#define BLOCK_SIZE 256	//should be a power of 2
#define SLM_SIZE 512
#if ((SLM_SIZE==16)||(SLM_SIZE==32)||(SLM_SIZE==64)||(SLM_SIZE==128)||(SLM_SIZE==256)||(SLM_SIZE==512)||(SLM_SIZE==1024)||(SLM_SIZE==2048))
#define SLMPOW2			//Uses bitwise modulu operations if the SLM size is a power of 2
#endif

////////////////////////////////////////////////////////////////////////////////
// forward declarations
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//Set correction parameters
////////////////////////////////////////////////////////////////////////////////
__global__ void uc2f(float *f, unsigned char *uc, int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx<N)
{
f[idx] = uc[idx]*2.0f*M_PI/256.0f - M_PI;
}
}