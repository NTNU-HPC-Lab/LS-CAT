#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, float *x, float *y) {
int index = threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < n; i += stride) {
y[i] = x[i] + y[i];
}
}