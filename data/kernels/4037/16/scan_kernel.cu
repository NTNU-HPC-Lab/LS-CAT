#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_kernel(unsigned int* d_bins, int size) {
int mid = threadIdx.x + blockDim.x * blockIdx.x;
if (mid >= size) return;

for (int s = 1; s <= size; s *= 2) {
int spot = mid - s;

unsigned int val = 0;
if (spot >= 0) val = d_bins[spot];
__syncthreads();
if (spot >= 0) d_bins[mid] += val;
__syncthreads();
}
}