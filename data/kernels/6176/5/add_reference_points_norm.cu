#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm){
unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;
unsigned int xIndex = blockIdx.x * blockDim.x + tx;
unsigned int yIndex = blockIdx.y * blockDim.y + ty;
__shared__ float shared_vec[16];
if (tx==0 && yIndex<height)
shared_vec[ty] = norm[yIndex];
__syncthreads();
if (xIndex<width && yIndex<height)
array[yIndex*pitch+xIndex] += shared_vec[ty];
}