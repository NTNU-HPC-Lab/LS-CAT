#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_distance_texture(hipTextureObject_t ref, int                 ref_width, float *             query, int                 query_width, int                 query_pitch, int                 height, float*              dist) {
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if ( xIndex<query_width && yIndex<ref_width) {
float ssd = 0.f;
for (int i=0; i<height; i++) {
float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
ssd += tmp * tmp;
}
dist[yIndex * query_pitch + xIndex] = ssd;
}
}