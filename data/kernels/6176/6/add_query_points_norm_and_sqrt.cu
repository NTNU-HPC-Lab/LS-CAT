#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
array[yIndex*pitch + xIndex] = sqrt(array[yIndex*pitch + xIndex] + norm[xIndex]);
}