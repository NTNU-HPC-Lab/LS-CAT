#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modified_insertion_sort(float * dist, int     dist_pitch, int *   index, int     index_pitch, int     width, int     height, int     k){

// Column position
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

// Do nothing if we are out of bounds
if (xIndex < width) {

// Pointer shift
float * p_dist  = dist  + xIndex;
int *   p_index = index + xIndex;

// Initialise the first index
p_index[0] = 0;

// Go through all points
for (int i=1; i<height; ++i) {

// Store current distance and associated index
float curr_dist = p_dist[i*dist_pitch];
int   curr_index  = i;

// Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
continue;
}

// Shift values (and indexes) higher that the current distance to the right
int j = min(i, k-1);
while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
--j;
}

// Write the current distance and index at their position
p_dist[j*dist_pitch]   = curr_dist;
p_index[j*index_pitch] = curr_index;
}
}
}