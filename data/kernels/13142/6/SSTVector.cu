#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SSTVector(float* V, int* addr, int N) {
int i = threadIdx.x;
if (i < N) {
int return_val = 0;
float element = V[i];
asm("/*");
asm("CPTX_BEGIN");
asm("sst.sstarr.f32 %0, [%1], %2, %3;" : "=r"(return_val) : "l"(&V[0]), "r"(i), "f"(element));
asm("CPTX_END");
asm("*/");
if (return_val != 0) *addr = return_val;
}
}