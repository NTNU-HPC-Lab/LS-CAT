#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SSTTest(float* V, int* addr, int N) {
int i = threadIdx.x;
if (i < N) {
int return_val = 0;
float element = V[i];
asm("/*");
asm("CPTX_BEGIN");
asm("sst.sstarr.f32 %0, [%1], %2, %3;" : "=r"(return_val) : "l"(&V[0]), "r"(i), "f"(element)); // perform SST instruction
asm("CPTX_END");
asm("*/");
if (return_val != 0) *addr = (int)(return_val - (intptr_t)&V[0])/4; // last thread stores the result
}
}