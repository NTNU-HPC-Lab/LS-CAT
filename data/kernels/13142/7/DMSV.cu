#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DMSV(float* M, float* V, float* R, int* addr, int N) {
int bid = blockIdx.x;
int tid = threadIdx.x;
if (tid < N) {
__shared__ float Vs[2*LENGTH];
for (int i = tid; i < tid+LENGTH; i+=BLOCK_SIZE) {
Vs[i] = V[i];
Vs[i+N] = V[i+N];
}
__syncthreads();

int numCols = (int)(*addr - (intptr_t)&V[0])/4;
float psum = 0.0;
for (int i = 0; i <= numCols; i++) {
int vid = (int)Vs[i+N];
//int vid = (int)V[i+N];
psum += M[N*vid + bid*BLOCK_SIZE + tid] * Vs[i];
//psum += M[N*vid + bid*BLOCK_SIZE + tid] * V[i];
}
R[bid*BLOCK_SIZE + tid] = psum;
}
}