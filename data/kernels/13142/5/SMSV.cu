#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SMSV(float* M, float* V, float* R, int* maddr, int* addr, int N) {
int tid = threadIdx.x;
if (tid < N) {
__shared__ int psum[LENGTH];
psum[tid] = 0; // initialize psum with 0s
__syncthreads(); // psum is finished being written to
int numCols = (int)(*addr - (intptr_t)&V[0])/4; // end of SST for vector
for (int i = 0; i <= numCols; i++) { // loop through columns
int vid = (int)V[i+N]; // vector index
int cEnd = (int)(maddr[vid] - (intptr_t)&M[2*N*vid])/4; // end of SST for column
if (tid <= cEnd) {
int mid = (int)M[2*N*vid + tid+N]; // matrix index
psum[mid] += M[2*N*vid + tid] * V[i];
}
__syncthreads();
}
R[tid] = psum[tid];
}
}