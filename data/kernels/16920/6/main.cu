#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cuSearchDoublet.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int *nSpM = NULL;
hipMalloc(&nSpM, XSIZE*YSIZE);
const float *spMmat = NULL;
hipMalloc(&spMmat, XSIZE*YSIZE);
const int *nSpB = NULL;
hipMalloc(&nSpB, XSIZE*YSIZE);
const float *spBmat = NULL;
hipMalloc(&spBmat, XSIZE*YSIZE);
const int *nSpT = NULL;
hipMalloc(&nSpT, XSIZE*YSIZE);
const float *spTmat = NULL;
hipMalloc(&spTmat, XSIZE*YSIZE);
const float *deltaRMin = NULL;
hipMalloc(&deltaRMin, XSIZE*YSIZE);
const float *deltaRMax = NULL;
hipMalloc(&deltaRMax, XSIZE*YSIZE);
const float *cotThetaMax = NULL;
hipMalloc(&cotThetaMax, XSIZE*YSIZE);
const float *collisionRegionMin = NULL;
hipMalloc(&collisionRegionMin, XSIZE*YSIZE);
const float *collisionRegionMax = NULL;
hipMalloc(&collisionRegionMax, XSIZE*YSIZE);
int *nSpMcomp = NULL;
hipMalloc(&nSpMcomp, XSIZE*YSIZE);
int *nSpBcompPerSpM_Max = NULL;
hipMalloc(&nSpBcompPerSpM_Max, XSIZE*YSIZE);
int *nSpTcompPerSpM_Max = NULL;
hipMalloc(&nSpTcompPerSpM_Max, XSIZE*YSIZE);
int *nSpBcompPerSpM = NULL;
hipMalloc(&nSpBcompPerSpM, XSIZE*YSIZE);
int *nSpTcompPerSpM = NULL;
hipMalloc(&nSpTcompPerSpM, XSIZE*YSIZE);
int *McompIndex = NULL;
hipMalloc(&McompIndex, XSIZE*YSIZE);
int *BcompIndex = NULL;
hipMalloc(&BcompIndex, XSIZE*YSIZE);
int *tmpBcompIndex = NULL;
hipMalloc(&tmpBcompIndex, XSIZE*YSIZE);
int *TcompIndex = NULL;
hipMalloc(&TcompIndex, XSIZE*YSIZE);
int *tmpTcompIndex = NULL;
hipMalloc(&tmpTcompIndex, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cuSearchDoublet<<<gridBlock,threadBlock>>>(nSpM,spMmat,nSpB,spBmat,nSpT,spTmat,deltaRMin,deltaRMax,cotThetaMax,collisionRegionMin,collisionRegionMax,nSpMcomp,nSpBcompPerSpM_Max,nSpTcompPerSpM_Max,nSpBcompPerSpM,nSpTcompPerSpM,McompIndex,BcompIndex,tmpBcompIndex,TcompIndex,tmpTcompIndex);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cuSearchDoublet<<<gridBlock,threadBlock>>>(nSpM,spMmat,nSpB,spBmat,nSpT,spTmat,deltaRMin,deltaRMax,cotThetaMax,collisionRegionMin,collisionRegionMax,nSpMcomp,nSpBcompPerSpM_Max,nSpTcompPerSpM_Max,nSpBcompPerSpM,nSpTcompPerSpM,McompIndex,BcompIndex,tmpBcompIndex,TcompIndex,tmpTcompIndex);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cuSearchDoublet<<<gridBlock,threadBlock>>>(nSpM,spMmat,nSpB,spBmat,nSpT,spTmat,deltaRMin,deltaRMax,cotThetaMax,collisionRegionMin,collisionRegionMax,nSpMcomp,nSpBcompPerSpM_Max,nSpTcompPerSpM_Max,nSpBcompPerSpM,nSpTcompPerSpM,McompIndex,BcompIndex,tmpBcompIndex,TcompIndex,tmpTcompIndex);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}