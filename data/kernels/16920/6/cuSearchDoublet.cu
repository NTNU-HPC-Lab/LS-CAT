#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSearchDoublet( const int* nSpM, const float* spMmat, const int* nSpB, const float* spBmat, const int* nSpT, const float* spTmat, const float* deltaRMin, const float* deltaRMax, const float* cotThetaMax, const float* collisionRegionMin, const float* collisionRegionMax, int* nSpMcomp, int* nSpBcompPerSpM_Max, int* nSpTcompPerSpM_Max, int* nSpBcompPerSpM, int* nSpTcompPerSpM, int* McompIndex, int* BcompIndex, int* tmpBcompIndex, int* TcompIndex, int* tmpTcompIndex) {
extern __shared__ float sharedMem[];
int* mPos = (int*)sharedMem;
int* isMcompat = (int*)&mPos[1];

if (threadIdx.x == 0) {
*isMcompat = false;
}
__syncthreads();

float rM = spMmat[blockIdx.x + (*nSpM) * 3];
float zM = spMmat[blockIdx.x + (*nSpM) * 2];

bool isBcompat(true);
bool isTcompat(true);

int offset(0);

while (offset < max(*nSpB, *nSpT)) {
isBcompat = true;

// Doublet search for bottom hits
if (threadIdx.x + offset < *nSpB) {
float rB = spBmat[threadIdx.x + offset + (*nSpB) * 3];
float zB = spBmat[threadIdx.x + offset + (*nSpB) * 2];

float deltaR = rM - rB;
if (deltaR > *deltaRMax) {
isBcompat = false;
}

if (deltaR < *deltaRMin) {
isBcompat = false;
}

float cotTheta = (zM - zB) / deltaR;
if (fabsf(cotTheta) > *cotThetaMax) {
isBcompat = false;
}

float zOrigin = zM - rM * cotTheta;
if (zOrigin < *collisionRegionMin || zOrigin > *collisionRegionMax) {
isBcompat = false;
}

if (isBcompat == true) {
int bPos = atomicAdd(&nSpBcompPerSpM[blockIdx.x], 1);
tmpBcompIndex[bPos + (*nSpB) * blockIdx.x] = threadIdx.x + offset;
}
}

isTcompat = true;

// Doublet search for top hits
if (threadIdx.x + offset < *nSpT) {
float rT = spTmat[threadIdx.x + offset + (*nSpT) * 3];
float zT = spTmat[threadIdx.x + offset + (*nSpT) * 2];
float deltaR = rT - rM;
if (deltaR < *deltaRMin) {
isTcompat = false;
}

if (deltaR > *deltaRMax) {
isTcompat = false;
}

if (isTcompat == true) {
float cotTheta = (zT - zM) / deltaR;
if (fabsf(cotTheta) > *cotThetaMax) {
isTcompat = false;
}

float zOrigin = zM - rM * cotTheta;
if (zOrigin < *collisionRegionMin || zOrigin > *collisionRegionMax) {
isTcompat = false;
}
}

if (isTcompat == true) {
int tPos = atomicAdd(&nSpTcompPerSpM[blockIdx.x], 1);
tmpTcompIndex[tPos + (*nSpT) * blockIdx.x] = threadIdx.x + offset;
}
}

offset += blockDim.x;
}

__syncthreads();

if (threadIdx.x == 0) {
if (nSpBcompPerSpM[blockIdx.x] > 0 && nSpTcompPerSpM[blockIdx.x] > 0) {
*mPos = atomicAdd(nSpMcomp, 1);
*isMcompat = true;
McompIndex[*mPos] = blockIdx.x;

int bMax = atomicMax(nSpBcompPerSpM_Max, nSpBcompPerSpM[blockIdx.x]);
int tMax = atomicMax(nSpTcompPerSpM_Max, nSpTcompPerSpM[blockIdx.x]);
}
}

__syncthreads();

if (*isMcompat == true) {
offset = 0;
while (offset <
max(nSpBcompPerSpM[blockIdx.x], nSpTcompPerSpM[blockIdx.x])) {
if (threadIdx.x + offset < nSpBcompPerSpM[blockIdx.x]) {
BcompIndex[threadIdx.x + offset + (*nSpB) * (*mPos)] =
tmpBcompIndex[threadIdx.x + offset + (*nSpB) * blockIdx.x];
}

if (threadIdx.x + offset < nSpTcompPerSpM[blockIdx.x]) {
TcompIndex[threadIdx.x + offset + (*nSpT) * (*mPos)] =
tmpTcompIndex[threadIdx.x + offset + (*nSpT) * blockIdx.x];
}
offset += blockDim.x;
}
}
}