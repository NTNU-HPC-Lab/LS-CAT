#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_mask_kernel(int n,  float *x, float mask_num, float *mask, float scale)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n && mask[i] == mask_num) x[i] *= scale;
}