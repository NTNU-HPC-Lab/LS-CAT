#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
int start = 0;
int end = size;

while (end - start > 0) {
int mid = start + (end - start) / 2;

float midVal = dist[mid];
if (midVal < val) {
start = mid + 1;
} else {
end = mid;
}
}

if (start == size) {
// No probability mass or precision problems; just return the
// first element
start = 0;
}

return start;
}
__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
// At the moment, each warp computes one sample value in the binary
// search due to divergence. It seems possible to compute multiple
// values and limit divergence though later on. However, no matter
// what, all block threads must participate in the hiprand_uniform
// call to update the generator state.

// The block determines the distribution for which we generate a point
for (long curDist = blockIdx.x;
curDist < distributions;
curDist += gridDim.x) {
for (int sampleBase = 0;
sampleBase < totalSamples; sampleBase += blockDim.y) {
// The warp determines the sample
int sample = sampleBase + threadIdx.y;

// All threads participate in this
float r = hiprand_uniform(&state[blockIdx.x]);

if (threadIdx.x == 0 && sample < totalSamples) {
// Find the bucket that a uniform sample lies in
int choice = binarySearchForMultinomial(
normDistPrefixSum + curDist * categories,
categories,
r);

// Torch indices are 1-based
dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
}
}
}
}