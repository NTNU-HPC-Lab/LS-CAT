#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
int start = 0;
int end = size;

while (end - start > 0) {
int mid = start + (end - start) / 2;

float midVal = dist[mid];
if (midVal < val) {
start = mid + 1;
} else {
end = mid;
}
}

if (start == size) {
// No probability mass or precision problems; just return the
// first element
start = 0;
}

return start;
}
__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
// At the moment, each warp computes one sample value in the binary
// search due to divergence. It seems possible to compute multiple
// values and limit divergence though later on. However, no matter
// what, all block threads must participate in the hiprand_uniform
// call to update the generator state.

// The block and warp determines the distribution for which we
// generate a point
for (long curDistBase = blockIdx.x * blockDim.y;
curDistBase < distributions;
curDistBase += gridDim.x * blockDim.y) {
// The warp determines the distribution
long curDist = curDistBase + threadIdx.y;

// All threads must participate in this
float r = hiprand_uniform(&state[blockIdx.x]);

if (threadIdx.x == 0 && curDist < distributions) {
// Find the bucket that a uniform sample lies in
int choice = binarySearchForMultinomial(
normDistPrefixSum + curDist * categories,
categories,
r);

// Torch indices are 1-based
dest[curDist * totalSamples + sample] = (float) choice + 1.0f;

// Without replacement, so update the original probability so it
// is not considered a second time
origDist[curDist * categories + choice] = 0.0f;
}
}
}