#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int DeviceDefaultStep() {
return gridDim.x * blockDim.x;
}
__device__ int DeviceDefaultIndex() {
return blockIdx.x * blockDim.x + threadIdx.x;
}
__global__ void KernelSelfPlusIters(const bool *indexers, int *iters, int count) {
int index = DeviceDefaultIndex();
int step = DeviceDefaultStep();
for (int i = index; i < count; i += step) {
if (indexers[i]) {
++iters[i];
}
}
}