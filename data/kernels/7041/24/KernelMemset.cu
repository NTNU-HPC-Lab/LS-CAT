#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int DeviceDefaultStep() {
return gridDim.x * blockDim.x;
}
__device__ int DeviceDefaultIndex() {
return blockIdx.x * blockDim.x + threadIdx.x;
}
__global__ void KernelMemset(bool *p, int len, bool value) {
int index = DeviceDefaultIndex();
int step = DeviceDefaultStep();
for (int i = index; i < len; i+= step) {
p[i] = value;
}
}