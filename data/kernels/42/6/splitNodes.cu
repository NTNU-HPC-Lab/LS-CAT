#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void splitNodes(int* octree, int* numNodes, int poolSize, int startNode) {

int index = blockIdx.x * blockDim.x + threadIdx.x;

//Don't do anything if its out of bounds
if (index < poolSize) {
int node = octree[2 * (index+startNode)];

//Split the node if its flagged
if (node & 0x40000000) {
//Get a new node tile
int newNode = atomicAdd(numNodes, 8);

//Point this node at the new tile
octree[2 * (index+startNode)] = (octree[2 * (index+startNode)] & 0xC0000000) | (newNode & 0x3FFFFFFF);

//Initialize new child nodes to 0's
for (int off = 0; off < 8; off++) {
octree[2 * (newNode + off)] = 0;
octree[2 * (newNode + off) + 1] = 0;
}
}
}

}