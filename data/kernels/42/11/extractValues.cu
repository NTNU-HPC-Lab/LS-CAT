#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extractValues(void* fb, int* voxels, int num_voxels, int* values) {
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < num_voxels) {
//TODO: Make this support other storage_type's besides int32
float* tile = (float*)fb;
values[index] = __float_as_int(tile[voxels[index]]);
}
}