#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
Y += blockIdx.x * dim;
dY += blockIdx.x * dim;
dX += blockIdx.x * dim;
const int idx = threadIdx.x;
__shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
float tmp;

// A two-level reduction to compute the inner products.
tmp = 0;
for (int i = idx; i < dim; i += blockDim.x) {
tmp += dY[i] * Y[i];
}
reduction_buffer[idx] = tmp;
__syncthreads();
if (idx == 0) {
tmp = reduction_buffer[0];
for (int i = 1; i < blockDim.x; ++i)
tmp += reduction_buffer[i];
reduction_buffer[0] = tmp;
}
__syncthreads();
// Compute gradient.
tmp = reduction_buffer[0];
for (int i = idx; i < dim; i += blockDim.x) {
dX[i] = Y[i] * (dY[i] - tmp);
}
}