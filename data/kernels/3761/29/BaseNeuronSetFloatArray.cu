#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronSetFloatArray(float *arr, int n_elem, int step, float val)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr[array_idx*step] = val;
}
}