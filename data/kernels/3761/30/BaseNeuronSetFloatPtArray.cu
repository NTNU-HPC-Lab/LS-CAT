#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronSetFloatPtArray(float *arr, int *pos, int n_elem, int step, float val)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr[pos[array_idx]*step] = val;
}
}