#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronGetIntPtArray(int *arr1, int *arr2, int *pos, int n_elem, int step1, int step2)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr2[array_idx*step2] = arr1[pos[array_idx]*step1];
}
}