#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetFloatArray(float *arr, int n_elem, int step, float val)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr[array_idx*step] = val;
}
}