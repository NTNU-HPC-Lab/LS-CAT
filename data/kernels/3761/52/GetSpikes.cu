#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetSpikes(double *spike_array, int array_size, int n_port, int n_var, float *port_weight_arr, int port_weight_arr_step, int port_weight_port_step, float *port_input_arr, int port_input_arr_step, int port_input_port_step)
{
int i_target = blockIdx.x*blockDim.x+threadIdx.x;
int port = blockIdx.y*blockDim.y+threadIdx.y;

if (i_target < array_size && port<n_port) {
int i_array = port*array_size + i_target;
int port_input = i_target*port_input_arr_step
+ port_input_port_step*port;
int port_weight = i_target*port_weight_arr_step
+ port_weight_port_step*port;
double d_val = (double)port_input_arr[port_input]
+ spike_array[i_array]
* port_weight_arr[port_weight];

port_input_arr[port_input] = (float)d_val;
}
}