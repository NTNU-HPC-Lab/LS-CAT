#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateExtNeuron(float *port_input_pt, float *port_value_pt, int n_node, int n_var, int n_port_var, int n_port)
{
int i_thread = threadIdx.x + blockIdx.x * blockDim.x;
if (i_thread<n_node*n_port) {
int i_port = i_thread%n_port;
int i_node = i_thread/n_port;
float *pip = port_input_pt + i_node*n_var + n_port_var*i_port;
//printf("port %d node %d pip %f\n", i_port, i_node, *pip);
port_value_pt[i_node*n_var + n_port_var*i_port]
= *pip;
*pip = 0.0;
}
}