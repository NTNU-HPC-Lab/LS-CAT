#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetupPoissKernel(hiprandState *curand_state, uint64_t n_dir_conn, unsigned long long seed)
{
uint64_t blockId   = (uint64_t)blockIdx.y * gridDim.x + blockIdx.x;
uint64_t i_conn = blockId * blockDim.x + threadIdx.x;
if (i_conn<n_dir_conn) {
hiprand_init(seed, i_conn, 0, &curand_state[i_conn]);
}
}