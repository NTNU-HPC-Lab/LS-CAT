#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
int blockID = blockIdx.x;
int threadID = threadIdx.x;
int blockOffset = blockID * n;

extern __shared__ int temp[];
temp[2 * threadID] = input[blockOffset + (2 * threadID)];
temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

int offset = 1;
for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
{
__syncthreads();
if (threadID < d)
{
int ai = offset * (2 * threadID + 1) - 1;
int bi = offset * (2 * threadID + 2) - 1;
temp[bi] += temp[ai];
}
offset *= 2;
}
__syncthreads();


if (threadID == 0) {
sums[blockID] = temp[n - 1];
temp[n - 1] = 0;
}

for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
{
offset >>= 1;
__syncthreads();
if (threadID < d)
{
int ai = offset * (2 * threadID + 1) - 1;
int bi = offset * (2 * threadID + 2) - 1;
int t = temp[ai];
temp[ai] = temp[bi];
temp[bi] += t;
}
}
__syncthreads();

output[blockOffset + (2 * threadID)] = temp[2 * threadID];
output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}