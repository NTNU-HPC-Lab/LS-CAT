#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *output, int length, int *n) {
int blockID = blockIdx.x;
int threadID = threadIdx.x;
int blockOffset = blockID * length;

output[blockOffset + threadID] += n[blockID];
}