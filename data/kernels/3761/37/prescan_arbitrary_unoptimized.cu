#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int locate(int val, int *data, int n)
{
int i_left = 0;
int i_right = n-1;
int i = (i_left+i_right)/2;
while(i_right-i_left>1) {
if (data[i] > val) i_right = i;
else if (data[i]<val) i_left = i;
else break;
i=(i_left+i_right)/2;
}

return i;
}
__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
extern __shared__ int temp[];// allocated on invocation
int threadID = threadIdx.x;

if (threadID < n) {
temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
temp[2 * threadID + 1] = input[2 * threadID + 1];
}
else {
temp[2 * threadID] = 0;
temp[2 * threadID + 1] = 0;
}


int offset = 1;
for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
{
__syncthreads();
if (threadID < d)
{
int ai = offset * (2 * threadID + 1) - 1;
int bi = offset * (2 * threadID + 2) - 1;
temp[bi] += temp[ai];
}
offset *= 2;
}

if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
{
offset >>= 1;
__syncthreads();
if (threadID < d)
{
int ai = offset * (2 * threadID + 1) - 1;
int bi = offset * (2 * threadID + 2) - 1;
int t = temp[ai];
temp[ai] = temp[bi];
temp[bi] += t;
}
}
__syncthreads();

if (threadID < n) {
output[2 * threadID] = temp[2 * threadID]; // write results to device memory
output[2 * threadID + 1] = temp[2 * threadID + 1];
}
}