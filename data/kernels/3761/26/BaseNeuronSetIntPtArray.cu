#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronSetIntPtArray(int *arr, int *pos, int n_elem, int step, int val)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr[pos[array_idx]*step] = val;
}
}