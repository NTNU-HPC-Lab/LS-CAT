#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *output, int length, int *n1, int *n2) {
int blockID = blockIdx.x;
int threadID = threadIdx.x;
int blockOffset = blockID * length;

output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}