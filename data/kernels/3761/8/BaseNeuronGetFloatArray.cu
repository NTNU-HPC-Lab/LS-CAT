#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronGetFloatArray(float *arr1, float *arr2, int n_elem, int step1, int step2)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr2[array_idx*step2] = arr1[array_idx*step1];
}
}