#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronSetIntArray(int *arr, int n_elem, int step, int val)
{
int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
if (array_idx<n_elem) {
arr[array_idx*step] = val;
}
}