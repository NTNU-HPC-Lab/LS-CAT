#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetSpikes(double *spike_array, int array_size, int n_port, int n_var, float *port_weight_arr, int port_weight_arr_step, int port_weight_port_step, float *port_input_arr, int port_input_arr_step, int port_input_port_step)
{
int i_array = threadIdx.x + blockIdx.x * blockDim.x;
if (i_array < array_size*n_port) {
int i_target = i_array % array_size;
int port = i_array / array_size;
int port_input = i_target*port_input_arr_step
+ port_input_port_step*port;
int port_weight = i_target*port_weight_arr_step
+ port_weight_port_step*port;
double d_val = (double)port_input_arr[port_input]
+ spike_array[i_array]
* port_weight_arr[port_weight];

port_input_arr[port_input] = (float)d_val;
}
}