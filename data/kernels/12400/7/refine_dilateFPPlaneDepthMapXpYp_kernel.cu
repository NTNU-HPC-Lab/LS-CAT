#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if((x + xp >= 0) && (y + yp >= 0) && (x + xp < width) && (y + yp < height) && (x < width) && (y < height))
{
float depth = maskMap[y * maskMap_p + x];
if(depth > 0.0f)
{
fpPlaneDepthMap[(y + yp) * fpPlaneDepthMap_p + (x + xp)] = fpPlaneDepth;
};
};
}