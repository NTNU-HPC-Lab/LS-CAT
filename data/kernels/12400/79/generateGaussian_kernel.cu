#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
int x = threadIdx.x - radius;
og[threadIdx.x] = __expf(-(x * x) / (2 * delta * delta));
}