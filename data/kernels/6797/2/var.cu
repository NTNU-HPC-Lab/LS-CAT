#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void var(int *a,int *b,int n,float mean)
{


int block=256*blockIdx.x;
float sum=0;


for(int i=block;i<min(block+256,n);i++)
{


sum=sum+(a[i]-mean)*(a[i]-mean);


}
b[blockIdx.x]=sum;

}