#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepare_indices(const unsigned num_keys, unsigned *data) {
unsigned index = threadIdx.x +
blockIdx.x * blockDim.x +
blockIdx.y * blockDim.x * gridDim.x;
if (index < num_keys) {
data[index] = index;
}
}