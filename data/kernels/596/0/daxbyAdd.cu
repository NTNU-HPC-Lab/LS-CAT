#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Compile: nvcc -o saxby saxby.cu
* Run: ./saxby
*/
__global__ void daxbyAdd(const float *A, const float *B, float *C, float x,int numElements){
int i = blockDim.x * blockIdx.x + threadIdx.x;
if(i < numElements){
C[i] = A[i]* x + B[i];
}
}