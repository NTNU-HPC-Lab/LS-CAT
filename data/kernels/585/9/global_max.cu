#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_max( int *d_values, int *d_global_max ) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int val = d_values[tid];
atomicMax(d_global_max, val);
}