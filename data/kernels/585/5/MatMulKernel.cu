#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
// Thread row and column within matrix
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

// Each thread computes one element of P
// by accumulating results into Pvalue
float Pvalue = 0;

// Multiply M and N
for (int k = 0; k < width; ++k) {
float Melement = *(Md + row*width + k);
float Nelement = *(Nd + k*width + col);
Pvalue += Melement * Nelement;
}

// Write Pvalue to device memory
// Each thread writes one element
*(Pd + row*width + col) = Pvalue;
}