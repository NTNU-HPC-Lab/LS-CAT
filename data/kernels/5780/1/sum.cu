#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int* input, int* sumOut) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
for(int j = 0; j < 100/(blockDim.x*gridDim.x); j++){
if (i < 100){
atomicAdd(sumOut, input[i+(j*blockDim.x*gridDim.x)]);
printf("NUM:%d Thread: %d ||\n",input[i+(j*blockDim.x*gridDim.x)],i);
}
}
__syncthreads();
}