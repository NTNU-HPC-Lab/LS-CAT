#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void max(int* input, int* maxOut) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
for(int j = 0; j < 100/(blockDim.x*gridDim.x); j++){
if (i < 100){
atomicMax(maxOut, input[i+(j*blockDim.x*gridDim.x)]);
printf("NUM:%d Thread: %d ||\n",input[i+(j*blockDim.x*gridDim.x)],i);
}
}
__syncthreads();
}