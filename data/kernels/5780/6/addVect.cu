#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVect(int *vect1 ,int *vect2 , int *resultVect){
int i = threadIdx.x + blockDim.x * blockIdx.x;
// printf("Thread id == %d || Block Id == %d\n",threadIdx.x,blockDim.x);
resultVect[i] = vect1[i] + vect2[i];
}