#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SoftmaxLossBackprop( const float *label, int num_labels, int batch_size, float *diff ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if( idx >= batch_size ) {
return;
}

const int label_value = static_cast<int>(label[ idx ]);

// For each item in the batch, decrease the result of the label's value by 1
diff[ idx * num_labels + label_value ] -= 1.0f;
}