#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FillOnes( float *vec, int size ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if( idx >= size ) {
return;
}

vec[ idx ] = 1.0f;
}