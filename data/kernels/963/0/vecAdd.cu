#include "hip/hip_runtime.h"
#include "includes.h"



#define N 1000	// size of vectors

#define T 10000// number of threads per block


__global__ void vecAdd(int *A, int *B, int *C) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
C[i] = A[i] * 10 + B[i];
}