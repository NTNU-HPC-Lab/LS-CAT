#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swan_fast_fill_word( uint *ptr, int len ) {
int idx = threadIdx.x + blockDim.x * blockIdx.x;
if( idx<len) {
ptr[idx] = 0;
}
}