#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void fill( float4 *localbuf, float val, float4* ptr, int offset, int N ) {
int idx= blockDim.x * blockIdx.x + threadIdx.x;
if( idx < N ) {
float4 t = localbuf[ idx ];
t.x += val;
t.y += val;
t.z += val;
t.w += val;

ptr[ offset + idx ] = t;
}
}