#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swan_fast_fill( uint4 *ptr, int len ) {
int idx = threadIdx.x + blockDim.x * blockIdx.x;
if( idx<len) {
ptr[idx] = make_uint4( 0,0,0,0 );
}
}