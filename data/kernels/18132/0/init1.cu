#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init1(uint32_t *A, uint32_t size) {
uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
if(id < size) A[id] = 1;
}