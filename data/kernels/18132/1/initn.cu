#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initn(uint32_t *A, uint32_t size, uint32_t n) {
uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
if(id < size) A[id] = n;
}