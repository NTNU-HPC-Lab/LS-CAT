#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigma(float x) {
return x / (1 + ((x < 0) ? -x : x));
}
__global__ void calcAll(float *w, float *b, float *v, uint32_t *wo, uint32_t *lo, uint32_t *ls, uint32_t s) {

uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
if(id < s) {
uint32_t ln = 0;
while(id > lo[ln]) ln++;

float sum = 0;
for(uint32_t i = 0; i < ls[ln - 1]; i++) {
sum += w[wo[ln] + ((id - lo[ln]) * ls[ln - 1]) + i] * v[lo[ln - 1] + i];
}
v[id] = sigma(sum + b[id]);
}

}