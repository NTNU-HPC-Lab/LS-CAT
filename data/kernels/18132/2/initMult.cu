#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initMult(uint32_t *A, uint32_t size, uint32_t mult) {
uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
if(id < size) A[id] = id * mult;
}