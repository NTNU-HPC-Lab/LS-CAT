#include "hip/hip_runtime.h"
#include "includes.h"
/// ================================================================
///
/// Disclaimer:  IMPORTANT:  This software was developed at theNT
/// National Institute of Standards and Technology by employees of the
/// Federal Government in the course of their official duties.
/// Pursuant to title 17 Section 105 of the United States Code this
/// software is not subject to copyright protection and is in the
/// public domain.  This is an experimental system.  NIST assumes no
/// responsibility whatsoever for its use by other parties, and makes
/// no guarantees, expressed or implied, about its quality,
/// reliability, or any other characteristic.  We would appreciate
/// acknowledgement if the software is used.  This software can be
/// redistributed and/or modified freely provided that any derivative
/// works bear some notice that they are derived from it, and any
/// modified versions bear some notice that they have been modified.
///
/// ================================================================

// ================================================================
//
// Author: Timothy Blattner
// Date:   Wed Nov 30 12:36:40 2011 EScufftDoubleComplex
//
// Functions that execute on the graphics card for doing
// Vector computation.
//
// ================================================================


#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// ================================================================
__device__ bool checkDistance(volatile int *maxesRow, volatile int *maxesCol, int nMax, int curIdx, int width)
{
int row = curIdx / width;
int col = curIdx % width;
int j;
//double dist;
for (j = 0; j < nMax; j++)
{

if (maxesRow[j] == row && maxesCol[j] == col)
return false;

//		dist = distance(maxesRow[j], row, maxesCol[j], col);

//		if (dist < MIN_DISTANCE)
//			return false;


}

return true;
}
__device__ bool checkDistance(int *maxesRow, int *maxesCol, int nMax, int curIdx, int width)
{
int row = curIdx / width;
int col = curIdx % width;
int j;
//double dist;
for (j = 0; j < nMax; j++)
{
if (maxesRow[j] == row && maxesCol[j] == col)
return false;

//dist = distance(maxesRow[j], row, maxesCol[j], col);

//if (dist < MIN_DISTANCE)
//	return false;


}

return true;
}
__device__ double distance(int x1, int x2, int y1, int y2)
{
return ((double(x1-x2))*(double(x1-x2)))+
((double(y1-y2))*(double(y1-y2)));
}
__global__ void reduce_max_filter_main(double *g_idata, double *g_odata, int * max_idx, unsigned int width, unsigned int height, int blockSize, int *maxes, int nMax)
{
__shared__ int smaxesRow[10];
__shared__ int smaxesCol[10];
__shared__ int smaxesVal[10];
__shared__ double sdata[THREADS_PER_BLOCK];
__shared__ int idxData[THREADS_PER_BLOCK];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockSize) + tid;
unsigned int gridSize = blockSize*gridDim.x;
if (tid < nMax)
{
smaxesVal[tid] = maxes[tid];
smaxesRow[tid] = smaxesVal[tid] / width;
smaxesCol[tid] = smaxesVal[tid] % width;
}
__syncthreads();

double myMax = -INFINITY;
int myMaxIndex;
double val;

while (i < width * height)
{
val = g_idata[i];
if (myMax < val)
{
// compute distance . . .
if (checkDistance(smaxesRow, smaxesCol,
nMax, i, width))
{
myMax = val;
myMaxIndex = i;
}
}

if (i+blockSize < width * height)
{
val = g_idata[i+blockSize];
if (myMax < val)
{

if (checkDistance(smaxesRow, smaxesCol,
nMax, i+blockSize, width))
{
myMax = val;
myMaxIndex = i+blockSize;
}
}
}

i += gridSize;
}

sdata[tid] = myMax;
idxData[tid] = myMaxIndex;

__syncthreads();

if (blockSize >= 512)
{
if (tid < 256)
{
if (myMax < sdata[tid + 256])
{
if (checkDistance(smaxesRow, smaxesCol,
nMax, idxData[tid+256],
width))
{
sdata[tid] = myMax = sdata[tid+256];
idxData[tid] = idxData[tid+256];
}
}
}
__syncthreads();
}

if (blockSize >= 256)
{
if (tid < 128)
{
if (myMax < sdata[tid + 128])
{
if (checkDistance(smaxesRow, smaxesCol,
nMax, idxData[tid+128],
width))
{
sdata[tid] = myMax = sdata[tid+128];
idxData[tid] = idxData[tid+128];
}
}
}
__syncthreads();
}

if (blockSize >= 128)
{
if (tid <   64)
{
if(myMax < sdata[tid +   64])
{
if (checkDistance(smaxesRow, smaxesCol,
nMax, idxData[tid+64],
width))
{
sdata[tid] = myMax = sdata[tid+64];
idxData[tid] = idxData[tid+64];
}
}
}
__syncthreads();
}

volatile double *vdata = sdata;
volatile int *vidxData = idxData;

volatile int *vsmaxesRow = smaxesRow;
volatile int *vsmaxesCol = smaxesCol;

if (tid < 32)
{
if (blockSize >=  64)
if (myMax < vdata[tid + 32])
{
if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+32],
width))
{
vdata[tid] = myMax = vdata[tid+32];
vidxData[tid] = vidxData[tid+32];
}
}

if (blockSize >=  32)
if (myMax < vdata[tid + 16])
{

if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+16],
width))
{
vdata[tid] = myMax = vdata[tid+16];
vidxData[tid] = vidxData[tid+16];
}
}

if (blockSize >=  16)
if (myMax < vdata[tid +  8])
{
if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+8],
width))
{
vdata[tid] = myMax = vdata[tid+8];
vidxData[tid] = vidxData[tid+8];
}
}

if (blockSize >=    8)
if (myMax < vdata[tid +  4])
{
if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+4],
width))
{
vdata[tid] = myMax = vdata[tid+4];
vidxData[tid] = vidxData[tid+4];
}
}

if (blockSize >=    4)
if (myMax < vdata[tid+2])
{
if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+2],
width))
{
vdata[tid] = myMax = vdata[tid+2];
vidxData[tid] = vidxData[tid+2];
}
}

if (blockSize >=    2)
if (myMax < vdata[tid +  1])
{
if (checkDistance(vsmaxesRow, vsmaxesCol,
nMax, vidxData[tid+1],
width))
{
vdata[tid] = myMax = vdata[tid+1];
vidxData[tid] = vidxData[tid+1];
}
}
__syncthreads();
}

if (tid == 0)
{
g_odata[blockIdx.x] = sdata[0];
max_idx[blockIdx.x] = idxData[0];

if (gridDim.x == 1)
maxes[nMax] = idxData[0];
}
}