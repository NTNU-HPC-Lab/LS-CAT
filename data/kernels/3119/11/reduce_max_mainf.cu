#include "hip/hip_runtime.h"
#include "includes.h"
/// ================================================================
///
/// Disclaimer:  IMPORTANT:  This software was developed at theNT
/// National Institute of Standards and Technology by employees of the
/// Federal Government in the course of their official duties.
/// Pursuant to title 17 Section 105 of the United States Code this
/// software is not subject to copyright protection and is in the
/// public domain.  This is an experimental system.  NIST assumes no
/// responsibility whatsoever for its use by other parties, and makes
/// no guarantees, expressed or implied, about its quality,
/// reliability, or any other characteristic.  We would appreciate
/// acknowledgement if the software is used.  This software can be
/// redistributed and/or modified freely provided that any derivative
/// works bear some notice that they are derived from it, and any
/// modified versions bear some notice that they have been modified.
///
/// ================================================================

// ================================================================
//
// Author: Timothy Blattner
// Date:   Wed Nov 30 12:36:40 2011 EScufftDoubleComplex
//
// Functions that execute on the graphics card for doing
// Vector computation.
//
// ================================================================


#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// ================================================================
__global__ void reduce_max_mainf(float *g_idata, float *g_odata, int * max_idx, unsigned int n, int blockSize)
{
__shared__ float sdata[THREADS_PER_BLOCK];
__shared__ int idxData[THREADS_PER_BLOCK];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockSize) + tid;
unsigned int gridSize = blockSize*gridDim.x;


float myMax = 0.0;
int myMaxIndex;
float val;

while (i < n)
{
val = g_idata[i];
if (myMax < val)
{
myMax = val;
myMaxIndex = i;
}

if (i+blockSize < n)
{
val = g_idata[i+blockSize];
if (myMax < val)
{
myMax = val;
myMaxIndex = i+blockSize;
}
}

i += gridSize;
}

sdata[tid] = myMax;
idxData[tid] = myMaxIndex;

__syncthreads();

if (blockSize >= 512)
{
if (tid < 256)
{
if (myMax < sdata[tid + 256])
{
sdata[tid] = myMax = sdata[tid+256];
idxData[tid] = idxData[tid+256];
}
}
__syncthreads();
}

if (blockSize >= 256)
{
if (tid < 128)
{
if (myMax < sdata[tid + 128])
{
sdata[tid] = myMax = sdata[tid+128];
idxData[tid] = idxData[tid+128];
}
}
__syncthreads();
}

if (blockSize >= 128)
{
if (tid <   64)
{
if(myMax < sdata[tid +   64])
{
sdata[tid] = myMax = sdata[tid+64];
idxData[tid] = idxData[tid+64];
}
}
__syncthreads();
}

volatile float *vdata = sdata;
volatile int *vidxData = idxData;

if (tid < 32)
{
if (blockSize >=  64)
if (myMax < vdata[tid + 32])
{
vdata[tid] = myMax = vdata[tid+32];
vidxData[tid] = vidxData[tid+32];
}

if (blockSize >=  32)
if (myMax < vdata[tid + 16])
{
vdata[tid] = myMax = vdata[tid+16];
vidxData[tid] = vidxData[tid+16];
}

if (blockSize >=  16)
if (myMax < vdata[tid +  8])
{
vdata[tid] = myMax = vdata[tid+8];
vidxData[tid] = vidxData[tid+8];
}

if (blockSize >=    8)
if (myMax < vdata[tid +  4])
{
vdata[tid] = myMax = vdata[tid+4];
vidxData[tid] = vidxData[tid+4];
}

if (blockSize >=    4)
if (myMax < vdata[tid+2])
{
vdata[tid] = myMax = vdata[tid+2];
vidxData[tid] = vidxData[tid+2];
}

if (blockSize >=    2)
if (myMax < vdata[tid +  1])
{
vdata[tid] = myMax = vdata[tid+1];
vidxData[tid] = vidxData[tid+1];
}
__syncthreads();
}

if (tid == 0)
{
g_odata[blockIdx.x] = sdata[0];
max_idx[blockIdx.x] = idxData[0];
}
}