#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relax(int* U, int* F, int* d, size_t gSize, int* adjMat) {
int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

if (globalThreadId < gSize) {
if (F[globalThreadId]) {
for (int i = 0; i < gSize; i++) {
if(adjMat[globalThreadId*gSize + i] && i != globalThreadId && U[i]) {
atomicMin(&d[i], d[globalThreadId] + adjMat[globalThreadId * gSize + i]);
}
}
}
}
}