#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int* U, int* F, int* d, int startNode, size_t gSize) {
int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

if (globalThreadId < gSize) {
U[globalThreadId] = 1;
F[globalThreadId] = 0;
d[globalThreadId] = INT_MAX;
}

if(globalThreadId == 0) {
d[globalThreadId] = 0;
U[globalThreadId] = 0;
F[globalThreadId] = 1;
}
}