#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min(int* U, int* d, int* outDel, int* minOutEdges, size_t gSize, int useD) {
int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

int pos1 = 2*globalThreadId;
int pos2 = 2*globalThreadId + 1;
int val1, val2;
if(pos1 < gSize) {
val1 = minOutEdges[pos1] + (useD ? d[pos1] : 0);
if(pos2 < gSize) {
val2 = minOutEdges[pos2] + (useD ? d[pos2] : 0);

val1 = val1 <= 0 ? INT_MAX : val1;
val2 = val2 <= 0 ? INT_MAX : val2;
if(useD) {
val1 = U[pos1] ? val1 : INT_MAX;
val2 = U[pos2] ? val2 : INT_MAX;
}
if(val1 > val2) {
outDel[globalThreadId] = val2;
}
else{
outDel[globalThreadId] = val1;
}
}
else {
val1 = val1 <= 0 ? INT_MAX : val1;
if(useD) {
val1 = U[pos1] ? val1 : INT_MAX;
}
outDel[globalThreadId] = val1;
}
}
}