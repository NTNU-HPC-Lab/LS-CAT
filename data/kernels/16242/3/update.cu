#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update(int* U, int* F, int* d, int* del, size_t gSize) {
int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

if (globalThreadId < gSize) {
F[globalThreadId] = 0;
if(U[globalThreadId] && d[globalThreadId] < del[0]) {
U[globalThreadId] = 0;
F[globalThreadId] = 1;
}
}
}