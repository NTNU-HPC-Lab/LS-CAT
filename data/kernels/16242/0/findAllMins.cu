#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findAllMins(int* adjMat, int* outVec, size_t gSize) {
int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
int ind = globalThreadId * gSize;
int min = INT_MAX;

if(globalThreadId < gSize) {
for(int i = 0; i < gSize; i++) {
if(adjMat[ind + i] < min && adjMat[ind + i] > 0) {
min = adjMat[ind + i];
}
}
outVec[globalThreadId] = min;
}
}