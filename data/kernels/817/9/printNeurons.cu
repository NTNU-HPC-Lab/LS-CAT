#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printNeurons(int *d_neuron_outputs, float *d_weighted_sums){
int neuron = blockIdx.x*blockDim.x + threadIdx.x;
printf("neuron: %d, weighted sum: %.2f, output: %d\n", neuron, d_weighted_sums[neuron], d_neuron_outputs[neuron]);
}