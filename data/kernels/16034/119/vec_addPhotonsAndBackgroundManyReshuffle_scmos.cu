#include "hip/hip_runtime.h"
#include "includes.h"








extern "C"




extern "C"






extern "C"





extern "C"





extern "C"




extern "C"


//=== Vector arithmetic ======================================================

extern "C"


extern "C"


extern "C"



extern "C"



extern "C"

extern "C"




//=== Vector-and-scalar arithmetic ===========================================

extern "C"


extern "C"


extern "C"


extern "C"




extern "C"


extern "C"


extern "C"


extern "C"









//=== Vector comparison ======================================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"



extern "C"




//=== Vector-and-scalar comparison ===========================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"


extern "C"











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"


// Calculate the arc sine of the input argument.
extern "C"


// Calculate the arc hyperbolic sine of the input argument.
extern "C"


// Calculate the arc tangent of the input argument.
extern "C"


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"


// Calculate the cube root of the input argument.
extern "C"


// Calculate ceiling of the input argument.
extern "C"


// Calculate the cosine of the input argument.
extern "C"


// Calculate the hyperbolic cosine of the input argument.
extern "C"


// Calculate the cosine of the input argument × p .
extern "C"


// Calculate the complementary error function of the input argument.
extern "C"


// Calculate the inverse complementary error function of the input argument.
extern "C"


// Calculate the scaled complementary error function of the input argument.
extern "C"


// Calculate the error function of the input argument.
extern "C"


// Calculate the inverse error function of the input argument.
extern "C"


// Calculate the base 10 exponential of the input argument.
extern "C"


// Calculate the base 2 exponential of the input argument.
extern "C"


// Calculate the base e exponential of the input argument.
extern "C"


// Calculate the base e exponential of the input argument, minus 1.
extern "C"


// Calculate the absolute value of its argument.
extern "C"


// Calculate the largest integer less than or equal to x.
extern "C"


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"


// Calculate the base 10 logarithm of the input argument.
extern "C"


// Calculate the value of l o g e ( 1 + x ) .
extern "C"


// Calculate the base 2 logarithm of the input argument.
extern "C"


// Calculate the doubleing point representation of the exponent of the input argument.
extern "C"


// Calculate the natural logarithm of the input argument.
extern "C"


// Calculate the standard normal cumulative distribution function.
extern "C"


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"


// Calculate reciprocal cube root function.
extern "C"


// Round input to nearest integer value in doubleing-point.
extern "C"


// Round to nearest integer value in doubleing-point.
extern "C"


// Calculate the reciprocal of the square root of the input argument.
extern "C"


// Calculate the sine of the input argument.
extern "C"


// Calculate the hyperbolic sine of the input argument.
extern "C"


// Calculate the sine of the input argument × p .
extern "C"


// Calculate the square root of the input argument.
extern "C"


// Calculate the tangent of the input argument.
extern "C"


// Calculate the hyperbolic tangent of the input argument.
extern "C"


// Calculate the gamma function of the input argument.
extern "C"


// Truncate input argument to the integral part.
extern "C"


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"

// Compute the positive difference between x and y.
extern "C"

// Divide two doubleing point values.
extern "C"

// Determine the maximum numeric value of the arguments.
extern "C"

// Determine the minimum numeric value of the arguments.
extern "C"

// Calculate the doubleing-point remainder of x / y.
extern "C"

// Calculate the square root of the sum of squares of two arguments.
extern "C"

// Return next representable single-precision doubleing-point value afer argument.
extern "C"

// Calculate the value of first argument to the power of second argument.
extern "C"

// Compute single-precision doubleing-point remainder.
extern "C"















extern "C"









extern "C"



extern "C"





extern "C"




extern "C"




extern "C"




extern "C"



extern "C"



//WARNING : device_sum size should be gridDim.x
__global__ void vec_addPhotonsAndBackgroundManyReshuffle_scmos (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground, double * scmos)
{


//print("to do as previous function");



int idx = threadIdx.x + blockIdx.x * blockDim.x;
int idy = threadIdx.y + blockIdx.y * blockDim.y;
int id = idy * gridDim.x * blockDim.x + idx;

if (id < n)
{
int idPSF=id/sizeSubImage;
int idModel=idPSF/numberPSFperModel;
int idoffset=id%sizeSubImage;
int idposit=idPSF%numberPSFperModel;
int idreshuffle=idModel*sizeSubImage +idposit*sizeSubImage*(n/(sizeSubImage*numberPSFperModel))       +idoffset;
output[idreshuffle]=input[id]*photonAndBackground[idPSF*2]+photonAndBackground[idPSF*2+1]+scmos[id];

}

}