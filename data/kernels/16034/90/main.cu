#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "vec_computePSF_phaseN.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int n = XSIZE*YSIZE;
double *kx = NULL;
hipMalloc(&kx, XSIZE*YSIZE);
double *ky = NULL;
hipMalloc(&ky, XSIZE*YSIZE);
double *kz = NULL;
hipMalloc(&kz, XSIZE*YSIZE);
double *pupil = NULL;
hipMalloc(&pupil, XSIZE*YSIZE);
double *phase = NULL;
hipMalloc(&phase, XSIZE*YSIZE);
double dx = 1;
double dy = 1;
double dz = 1;
int *sparseIndexEvenDisk = NULL;
hipMalloc(&sparseIndexEvenDisk, XSIZE*YSIZE);
int *sparseIndexOddDisk = NULL;
hipMalloc(&sparseIndexOddDisk, XSIZE*YSIZE);
double *fft = NULL;
hipMalloc(&fft, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
vec_computePSF_phaseN<<<gridBlock,threadBlock>>>(n,kx,ky,kz,pupil,phase,dx,dy,dz,sparseIndexEvenDisk,sparseIndexOddDisk,fft);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
vec_computePSF_phaseN<<<gridBlock,threadBlock>>>(n,kx,ky,kz,pupil,phase,dx,dy,dz,sparseIndexEvenDisk,sparseIndexOddDisk,fft);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
vec_computePSF_phaseN<<<gridBlock,threadBlock>>>(n,kx,ky,kz,pupil,phase,dx,dy,dz,sparseIndexEvenDisk,sparseIndexOddDisk,fft);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}