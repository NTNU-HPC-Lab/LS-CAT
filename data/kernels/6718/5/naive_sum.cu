#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void warpReduce(volatile float *sdata, int tid, int bid, int size)
{
if (bid + 32 < size) sdata[tid] += sdata[tid + 32];
if (bid + 16 < size) sdata[tid] += sdata[tid + 16];
if (bid + 8 < size) sdata[tid] += sdata[tid + 8];
if (bid + 4 < size) sdata[tid] += sdata[tid + 4];
if (bid + 2 < size) sdata[tid] += sdata[tid + 2];
if (bid + 1 < size) sdata[tid] += sdata[tid + 1];
}
__global__ void naive_sum(float *input, int size, float *out)
{
const unsigned int tid = threadIdx.x;
const unsigned int bid = blockIdx.x * blockDim.x * 2 + tid;
extern __shared__ float sdata[];
if (!(bid < size)) return;
sdata[tid] = input[bid];
if (bid + blockDim.x < size) sdata[tid] += input[bid + blockDim.x];
__syncthreads();
for (unsigned int offset = blockDim.x/2; offset > 32; offset /= 2) {
if (tid < offset && bid + offset < size) sdata[tid] += sdata[tid + offset];
__syncthreads();
}
if (tid < 32) warpReduce(sdata, tid, bid, size);
if (tid == 0) out[blockIdx.x] = sdata[0];
}