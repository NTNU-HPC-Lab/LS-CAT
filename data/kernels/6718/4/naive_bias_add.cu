#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_bias_add(float *in, int size, float *bias, int bias_size)
{
int bid = blockIdx.x * blockDim.x + threadIdx.x;
if (!(bid < size)) return;
int bias_offset = bid - (bid / bias_size) * bias_size;
in[bid] += bias[bias_offset];
}