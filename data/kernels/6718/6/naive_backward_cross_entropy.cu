#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_backward_cross_entropy(float *in, int *one_hot_classes, float batches, int size, float *out)
{
int bid = blockIdx.x * blockDim.x + threadIdx.x;
if (!(bid < size)) return;
out[bid] = (in[bid] - one_hot_classes[bid]) / batches;
}