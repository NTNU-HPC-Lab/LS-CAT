#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_matrix_transpose(float *input, int axis_0, int axis_1, float *output)
{
__shared__ float tile[TILE_DIM][TILE_DIM + 1];

int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
for (int i = 0; i < TILE_DIM && y + i < axis_1 && x < axis_0; i += BLOCK_HEIGHT) {
tile[threadIdx.y + i][threadIdx.x] = input[(y + i) * axis_0 + x];
}

__syncthreads();

x = blockIdx.y * TILE_DIM + threadIdx.x;
y = blockIdx.x * TILE_DIM + threadIdx.y;

for (int i = 0; i < TILE_DIM && y + i < axis_1 && x < axis_0; i += BLOCK_HEIGHT) {
output[(y + i) * axis_0 + x] = tile[(threadIdx.x)][threadIdx.y + i];
}
}