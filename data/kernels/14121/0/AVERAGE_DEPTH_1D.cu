#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel Average with Depth
extern "C"

//Converting 2D coordinates into one 1D coordinate
__global__ void AVERAGE_DEPTH_1D(int envSizeX, int envSizeY, float* envData, int depth){
int tidX = blockIdx.x * blockDim.x + threadIdx.x;
int tidY = blockIdx.y * blockDim.y + threadIdx.y;

float moyenne = 0;
int nbNombre = 0;

if(tidX < envSizeX && tidY < envSizeY){
for(int l = tidX - depth; l <= tidX + depth; l++){
if(l < 0){
int ltemp = l;
ltemp += envSizeX;

for(int k = tidY - depth; k <= tidY + depth; k++){
if(k < 0){
int ktemp = k;
ktemp += envSizeY;
if(envData[envSizeX * ltemp + ktemp] != -1){
moyenne += envData[envSizeX * ltemp + ktemp];
nbNombre++;
}
}
else if(k > envSizeY - 1){
int ktemp = k;
ktemp -= envSizeY;
if(envData[envSizeX * ltemp + ktemp] != -1){
moyenne += envData[envSizeX * ltemp + ktemp];
nbNombre++;
}
}
else{
if(envData[envSizeX * ltemp + k] != -1){
moyenne += envData[envSizeX * ltemp + k];
nbNombre++;
}
}
}
}
else if(l > envSizeX - 1){
int ltemp = l;
ltemp -= envSizeX;

for(int k = tidY - depth; k <= tidY + depth; k++){
if(k < 0){
int ktemp = k;
ktemp += envSizeY;
if(envData[envSizeX * ltemp + ktemp] != -1){
moyenne += envData[envSizeX * ltemp + ktemp];
nbNombre++;
}
}
else if(k > envSizeY - 1){
int ktemp = k;
ktemp -= envSizeY;
if(envData[envSizeX * ltemp + ktemp] != -1){
moyenne += envData[envSizeX * ltemp + ktemp];
nbNombre++;
}
}
else{
if(envData[envSizeX * ltemp + k] != -1){
moyenne += envData[envSizeX * ltemp + k];
nbNombre++;
}
}
}
}
else{
for(int k = tidY - depth; k <= tidY + depth; k++){
if(k < 0){
int ktemp = k;
ktemp += envSizeY;
if(envData[envSizeX * l + ktemp] != -1){
moyenne += envData[envSizeX * l + ktemp];
nbNombre++;
}
}
else if(k > envSizeY - 1){
int ktemp = k;
ktemp -= envSizeY;
if(envData[envSizeX * l + ktemp] != -1){
moyenne += envData[envSizeX * l + ktemp];
nbNombre++;
}
}
else{
if(envData[envSizeX * l + k] != -1){
moyenne += envData[envSizeX * l + k];
nbNombre++;
}
}
}
}
}
if(nbNombre != 0){
envData[envSizeX * tidX + tidY] = moyenne / nbNombre;
}
}
__syncthreads();
}