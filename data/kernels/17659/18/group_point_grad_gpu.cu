#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
int index = threadIdx.x;
idx += m*nsample*index;
grad_out += m*nsample*c*index;
grad_points += n*c*index;

for (int j=0;j<m;++j) {
for (int k=0;k<nsample;++k) {
int ii = idx[j*nsample+k];
for (int l=0;l<c;++l) {
grad_points[ii*c+l] += grad_out[j*nsample*c+k*c+l];
}
}
}
}