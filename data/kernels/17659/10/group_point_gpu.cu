#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
for (int i=0;i<b;++i) {
for (int j=0;j<m;++j) {
for (int k=0;k<nsample;++k) {
int ii = idx[j*nsample+k];
for (int l=0;l<c;++l) {
out[j*nsample*c+k*c+l] = points[ii*c+l];
}
}
}
points+=n*c;
idx+=m*nsample;
out+=m*nsample*c;
}
}