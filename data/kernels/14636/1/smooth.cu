#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth( unsigned char *entrada,unsigned char *saida, int n_linhas, int n_colunas ) {
//Calcula a posição no vetor (id_bloco * total_blocos + id_thread)
int posicao = blockIdx.x * blockDim.x + threadIdx.x;
//Se a posição não é maior que o limite da imagem original...
if(posicao < (n_linhas)*(n_colunas)) {
//soma o valor da região 5x5 em torno no pixel
saida[posicao] =entrada[posicao]+
entrada[posicao+(n_colunas+4)]+
entrada[posicao+(2*(n_colunas+4))]+
entrada[posicao+(3*(n_colunas+4))]+
entrada[posicao+(4*(n_colunas+4))]+
entrada[posicao+1]+
entrada[posicao+(n_colunas+4)+1]+
entrada[posicao+(2*(n_colunas+4))+1]+
entrada[posicao+(3*(n_colunas+4))+1]+
entrada[posicao+(4*(n_colunas+4))+1]+
entrada[posicao+2]+
entrada[posicao+(n_colunas+4)+2]+
entrada[posicao+(2*(n_colunas+4))+2]+
entrada[posicao+(3*(n_colunas+4))+2]+
entrada[posicao+(4*(n_colunas+4))+2]+
entrada[posicao+3]+
entrada[posicao+(n_colunas+4)+3]+
entrada[posicao+(2*(n_colunas+4))+3]+
entrada[posicao+(3*(n_colunas+4))+3]+
entrada[posicao+(4*(n_colunas+4))+3]+
entrada[posicao+4]+
entrada[posicao+(n_colunas+4)+4]+
entrada[posicao+(2*(n_colunas+4))+4]+
entrada[posicao+(3*(n_colunas+4))+4]+
entrada[posicao+(4*(n_colunas+4))+4];
//calcula a média
saida[posicao] = saida[posicao]/25;
}
}