#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
error[i] = truth[i] ? -pred[i] : pred[i];
delta[i] = (truth[i] > 0) ? 1 : -1;
}
}