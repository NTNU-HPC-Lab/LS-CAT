#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Sum_V(const float* a, float* partial_sums, const int n)
{
extern __shared__ float sdata[];

int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;
int ti = threadIdx.x;

// move global input data to shared memory, pad with zeros
float x = 0.0f;
if (i < n)
{
x = a[i];
}
sdata[ti] = x;

__syncthreads();

// use parallel reduction to contiguously reduce to partial sums
for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
{
if (ti < offset)
{
sdata[ti] += sdata[ti + offset];
}

__syncthreads();
}

if (ti == 0)
{
partial_sums[blockIdx.x] = sdata[0];
}
}