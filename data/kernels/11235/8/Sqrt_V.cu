#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Sqrt_V(const float* a, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = sqrtf(a[i]);
}
}