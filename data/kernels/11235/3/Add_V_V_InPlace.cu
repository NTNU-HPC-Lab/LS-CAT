#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Add_V_V_InPlace(const float* a, int aOffset, float* b, int bOffset, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
b[i + bOffset] = a[i + aOffset] + b[i + bOffset];
}
}