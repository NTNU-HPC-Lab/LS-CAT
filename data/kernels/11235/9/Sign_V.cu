#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Sign_V(const float* a, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = copysignf(1.0f, a[i]);
}
}