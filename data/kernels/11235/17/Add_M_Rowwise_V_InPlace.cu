#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Add_M_Rowwise_V_InPlace(const float* a, const int rows, const int cols, const int cols2, float* b, const int n)
{
extern __shared__ float sdata[];

int rowsPerBlock = blockDim.x / cols;
int usedPerBlock = rowsPerBlock * cols;
int unusedPerBlock = blockDim.x - usedPerBlock;

int ti = threadIdx.x;
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x - (unusedPerBlock * blockId);
int ri = i / cols;
int riLocal = ri % rowsPerBlock;
int tiLocal = ti - riLocal * cols;
bool inData = i < n && ti < usedPerBlock;

float x = 0.0f;
if (inData)
{
x = a[i];
}
sdata[ti] = x;

__syncthreads();

// calculate each rows derivatives (in sdata) sum
for (int offset = cols2 / 2; offset > 0; offset >>= 1)
{
if (tiLocal < offset)
{
float other = (ti + offset) / cols == riLocal ? sdata[ti + offset] : 0.0f;

sdata[ti] = sdata[ti] + other;
}

__syncthreads();
}

if (tiLocal == 0)
{
b[ri] = b[ri] + sdata[riLocal * cols];
}
}