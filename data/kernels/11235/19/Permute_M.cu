#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Permute_M(const float* a, const float* permutedDimensions, const float* originalStrides, float* out, const float* permutedStrides, const int rank, const int n)
{
extern __shared__ float sdata[];

int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

float* bufferIndices = &sdata[threadIdx.x * rank * 2];
float* resultIndices = &bufferIndices[rank];

if (i < n)
{
int flatIndex = i;

for (int y = 0; y < rank; y++)
{
bufferIndices[y] = (int) (flatIndex / originalStrides[y]);
flatIndex -= bufferIndices[y] * originalStrides[y];
}

for (int y = 0; y < rank; y++)
{
resultIndices[y] = bufferIndices[(int) permutedDimensions[y]];
}

int permutedIndex = 0;

for (int y = 0; y < rank; y++)
{
permutedIndex += resultIndices[y] * permutedStrides[y];
}

out[permutedIndex] = a[i];
}
}