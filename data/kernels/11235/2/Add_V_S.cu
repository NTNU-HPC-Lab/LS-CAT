#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Add_V_S(const float* a, const float b, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = a[i] + b;
}
}