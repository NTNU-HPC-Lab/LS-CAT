#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void RepeatReshapeCopy_V_MRows(const float* a, float* b, const int rows, const int cols, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < cols)
{
float value = a[i];

while (i < n)
{
b[i] = value;

i += cols;
}
}
}