#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Softmax_Rowwise_M_Backward(const float* origin, const float* adjoint, const float* primal, const float* prevMaxs, const float* prevMaxIndices, const float* prevSums, float* out, const int rows, const int cols, const int cols2, const int n)
{
extern __shared__ float sdata[];
float* rowBuffer = sdata;
float* originData = &sdata[blockDim.x];
float* adjointData = &sdata[blockDim.x * 2];
float* primalData = &sdata[blockDim.x * 3];
float* outData = &sdata[blockDim.x * 4];

int rowsPerBlock = blockDim.x / cols;
int usedPerBlock = rowsPerBlock * cols;
int unusedPerBlock = blockDim.x - usedPerBlock;

int ti = threadIdx.x;
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x - (unusedPerBlock * blockId);
int ri = i / cols;
int riLocal = ri % rowsPerBlock;
int tiLocal = ti - riLocal * cols;
bool inData = i < n && ti < usedPerBlock;

float prevMax = prevMaxs[ri];
int prevMaxIndex = prevMaxIndices[ri];
float prevSum = prevSums[ri];

if (inData)
{
originData[ti] = origin[i];
adjointData[ti] = adjoint[i];
primalData[ti] = primal[i];
}

// Div_DM_D				DM (direct)						 D (indirect via Sum_DM)
rowBuffer[ti] = adjointData[ti] / prevSum + adjointData[ti] * (originData[ti] / (prevSum * prevSum));

// Exp_DM				DM (direct)
rowBuffer[ti] = rowBuffer[ti] * __expf(originData[ti] - prevMax);
outData[ti] = rowBuffer[ti];

__syncthreads();

// calculate each rows derivatives (in rowBuffer) sum
for (int offset = cols2 / 2; offset > 0; offset >>= 1)
{
if (tiLocal < offset)
{
float other = (ti + offset) / cols == riLocal ? rowBuffer[ti + offset] : 0.0f;

rowBuffer[ti] = rowBuffer[ti] + other;
}

__syncthreads();
}

// Item_DM		D (indirect via Max op via Sub_DM_D op (left part for DM is just passthrough of gradient, so nothing to do there))
if (tiLocal == prevMaxIndex)
{
outData[ti] = outData[ti] - rowBuffer[riLocal * cols];
}

if (inData)
{
out[i] = outData[ti];
}
}