#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Softmax_Rowwise_M(const float* a, float* maxPerRow, float* maxPerRowIndices, float* sumPerRow, const int rows, const int cols, const int cols2, float* out, const int n)
{
extern __shared__ float sdata[];
float* rowBuffer = &sdata[blockDim.x];

int rowsPerBlock = blockDim.x / cols;
int usedPerBlock = rowsPerBlock * cols;
int unusedPerBlock = blockDim.x - usedPerBlock;

int ti = threadIdx.x;
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x - (unusedPerBlock * blockId);
int ri = i / cols;
int riLocal = ri % rowsPerBlock;
int tiLocal = ti - riLocal * cols;
bool inData = i < n && ti < usedPerBlock;

float x = 0.0f;
if (inData)
{
x = a[i];
}
sdata[ti] = rowBuffer[ti] = x;

__syncthreads();

// find each rows max value
for (int offset = cols2 / 2; offset > 0; offset >>= 1)
{
if (tiLocal < offset)
{
float currentMax = rowBuffer[ti];
float other = (ti + offset) / cols == riLocal ? rowBuffer[ti + offset] : 0.0f;

rowBuffer[ti] = other > currentMax ? other : currentMax;
}

__syncthreads();
}

// subtract each value from that row's maximum
if (inData)
{
sdata[ti] = __expf(sdata[ti] - rowBuffer[riLocal * cols]);

if (tiLocal == 0)
{
maxPerRow[ri] = rowBuffer[riLocal * cols];
}
}
rowBuffer[ti] = sdata[ti];

__syncthreads();

// write out max index
if (maxPerRow[ri] == a[i])
{
maxPerRowIndices[ri] = tiLocal;
}

// calculate each rows sum
for (int offset = cols2 / 2; offset > 0; offset >>= 1)
{
if (tiLocal < offset)
{
float other = (ti + offset) / cols == riLocal ? rowBuffer[ti + offset] : 0.0f;

rowBuffer[ti] = rowBuffer[ti] + other;
}

__syncthreads();
}

if (inData)
{
out[i] = sdata[ti] / rowBuffer[riLocal * cols];

if (tiLocal == 0)
{
sumPerRow[ri] = rowBuffer[riLocal * cols];
}
}
}