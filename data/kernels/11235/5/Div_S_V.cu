#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Div_S_V(const float a, const float* b, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = a / b[i];
}
}