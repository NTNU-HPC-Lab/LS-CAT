#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Log_V(const float* a, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = __logf(a[i]);
}
}