#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Rel_V(const float* a, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = (fabsf(a[i]) + a[i]) / 2.0f;
}
}