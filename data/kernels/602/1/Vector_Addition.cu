#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition ( const int *dev_a , const int *dev_b , int *dev_c)
{
//Get the id of thread within a block
unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x ;

while ( tid < N ) // check the boundry condition for the threads
{
dev_c [tid] = dev_a[tid] + dev_b[tid] ;
tid+= blockDim.x * gridDim.x ;
}
}