#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transponer(float* entrada, float* salida, int ANCHO){
int tx = blockIdx.x*blockDim.x + threadIdx.x;
int ty = blockIdx.y*blockDim.y + threadIdx.y;
salida[tx*ANCHO + ty] = entrada[ty*ANCHO + tx];
}