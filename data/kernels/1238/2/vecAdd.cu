#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(int *a, int *b, int *c) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while(tid < D)
{
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}

}