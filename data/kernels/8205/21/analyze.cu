#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void analyze(const float *input, float *sum, int numElements) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < numElements) {
atomicAdd(sum + i, input[i]);
}
}