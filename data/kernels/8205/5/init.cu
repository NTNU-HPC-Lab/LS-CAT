#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
auto i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n)
v[i] = val;
if (i == 0)
printf("init\n");
}