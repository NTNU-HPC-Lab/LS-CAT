#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Multiplying a 2D matrix using CUDA
*/


#define BLOCK_SIZE 16


__global__ void gpu_matrix_mul( int *a, int *b, int *c, int m, int n, int k){
int row = blockIdx.y + blockDim.y * threadIdx.y;
int col = blockIdx.x + blockDim.x * threadIdx.x;
int sum = 0;

if(col < k && row < m){
for(int i = 0; i < n; i++){
sum += a[row*n + i] * b[i*k + col];
}
c[row * k + col] = sum;
}
}