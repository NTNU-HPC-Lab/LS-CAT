#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_norm_dVector_kernel(double *a, double *partial_sum, int n) {
__shared__ double partial_sums[BLOCKSIZE];

double local_sum = 0;

int id = blockIdx.x*blockDim.x + threadIdx.x;
int partial_index = threadIdx.x;

while (id < n) {
local_sum += (a[id] * a[id]);
id += (blockDim.x * gridDim.x); // this thread may have to handle multiple sums
}

partial_sums[partial_index] = local_sum;

__syncthreads();

int sum_level = blockDim.x >> 1; // divide by 2

while (sum_level != 0) {
if (partial_index < sum_level) {
partial_sums[partial_index] += partial_sums[partial_index + sum_level];
}

__syncthreads();

sum_level >>= 1; // divide by 2
}

if (partial_index == 0) {
// if we are the thread processing index 0 of partial_sums for our block
partial_sum[blockIdx.x] = partial_sums[0];
}
// at this point there is still some partial somes left to compute
// inefficient to do so on GPU. Let CPU do this
}