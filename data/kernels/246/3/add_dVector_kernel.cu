#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_dVector_kernel(double *a, double *b, double *c, int n) {
int id = blockIdx.x*blockDim.x + threadIdx.x;

if (id < n)
c[id] = a[id] + b[id];
}