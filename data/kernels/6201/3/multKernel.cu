#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] * b[i] * 100;
}