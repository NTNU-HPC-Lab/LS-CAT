#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blur(uchar3 *input, uchar3 *output,int width, int height) {

int matrix[7][7] = {{0,0,1,2,1,0,0},{0,3,13,22,13,3,0},{1,3,59,97,59,13,1},{2,22,97,159,97,22,2},{1,3,59,97,59,3,1},{0,3,13,22,13,3,0},{0,0,1,2,1,0,0}};


int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

//if ((gridDim.x * gridDim.y) < width * height){

int tid = y*width + x;

int outputTemp = 0;

int sommeCoef = 0;

if (x<width){

if (y<height){

if (x>3 && x<width-3 && y>3 && y<height-3){

for (int i=0; i<7; i++){

for (int j=0; j<7; j++){

outputTemp += input[(y-3+i)*width+(x-3+j)].x*matrix[j][i];

sommeCoef += matrix[j][i];

}


output[tid].x = outputTemp / sommeCoef;

output[tid].z = output[tid].y = output[tid].x;

}
}

}

}

// }
}