#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hsv2rgb(float *inputH, float *inputS, float *inputV, uchar3 *output, int width, int height) {


int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int tid = y*width + x;

if (x<width){

if (y<height){

float H = inputH[tid];

float S = inputS[tid];

float V = inputV[tid];

float d =inputH[tid]/60;

int hi = (int)d%6;

float f = d - hi;

float l = V * (1 - S);

float m = V * (1 - f*S);

float n = V * (1 - (1 - f)*S);


if ((H>=0)&&(H<60)){

output[tid].x = (int)(V*255);
output[tid].y =(int) (n*255);
output[tid].z = (int)(l*255);

}
if ((H>=60)&&(H<120)){

output[tid].x = (int)(m*255);
output[tid].y = (int)(V*255);
output[tid].z = (int)(l*255);

}
if ((H>=120)&&(H<180)){

output[tid].x = (int)(l*255);
output[tid].y = (int)(V*255);
output[tid].z = (int)(n*255);

}
if ((H>=180)&&(H<240)){

output[tid].x = (int)(l*255);
output[tid].y = (int)(m*255);
output[tid].z = (int)(V*255);

}
if ((H>=240)&&(H<300)){

output[tid].x = (int)(n*255);
output[tid].y = (int)(l*255);
output[tid].z = (int)(V*255);

}

if ((H>=300)&&(H<360)){

output[tid].x = (int)(V*255);
output[tid].y = (int)(l*255);
output[tid].z = (int)(m*255);

}


}
}
}