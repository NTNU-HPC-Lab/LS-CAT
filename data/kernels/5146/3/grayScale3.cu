#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale3(uchar3 *input, uchar3 *output,int width, int height) {


int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

//if ((gridDim.x * gridDim.y) < width * height){

int tid = y*width + x;

if (x<width){

if (y<height){


output[tid].x = (input[tid].x + input[tid].y +
input[tid].z) / 3;

output[tid].z = output[tid].y = output[tid].x;

}
}

// }
}