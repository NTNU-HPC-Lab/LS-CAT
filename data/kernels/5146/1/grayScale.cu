#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale(uchar3 *input, uchar3 *output) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
output[tid].x = (input[tid].x + input[tid].y +
input[tid].z) / 3;
output[tid].z = output[tid].y = output[tid].x;
}