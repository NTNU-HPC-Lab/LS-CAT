#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blendingGray(uchar3 *input, uchar3 *input2, uchar3 *output,int width, int height,float coefficient) {


int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;


int tid = y*width + x;

int nbPixels = width * height;
float prod = coefficient * (float) nbPixels;
int prodfin = (int) prod;


if (x<width){

if (y<height){

if (tid <= prodfin){

output[tid].x = input[tid].x;

output[tid].z = output[tid].y = output[tid].x;

}
else{

output[tid].x = input2[tid].x;

output[tid].z = output[tid].y = output[tid].x;

}

}
}


}