#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale2(uchar3 *input, uchar3 *output,int width, int height) {

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int w = blockDim.x * gridDim.x;

//if ((gridDim.x * gridDim.y) < width * height){

int tid = y*w + x;

output[tid].x = (input[tid].x + input[tid].y +
input[tid].z) / 3;
output[tid].z = output[tid].y = output[tid].x;

// }
}