#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime


// Helper functions and utilities to work with CUDA


#define N 256
//#define M 256


//__global__ÉùÃ÷µÄº¯Êý£¬¸æËß±àÒëÆ÷Õâ¶Î´úÂë½»ÓÉCPUµ÷ÓÃ£¬ÓÉGPUÖ´ÐÐ

__global__ void matrix_mult(float *dev_a, float* dev_b, float* dev_c, int Width)
{
int Row = blockIdx.y*blockDim.y+threadIdx.y;
int Col = blockIdx.x*blockDim.x+threadIdx.x;
if ((Row < Width) && (Col < Width)) {
float Pvalue = 0;
for (int k = 0; k < Width; k++)
{
Pvalue += dev_a[Row*Width + k] * dev_b[k*Width+Col];
}
dev_c[Row*Width + Col] = Pvalue;

}
}