#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2



// dp - cost aggregation array
// cost_image - m x n x D array
// d - use every d channels of input to conserve register memory
// m - image rows
// n - image columns
// D - depth
// depth_stride - pitch along depth dimension
// row_stride - pitch along row dimension


__global__ void argmin_3d_mat(float * dp, int * stereo_im, int m, int n)
{
int col = blockDim.x * blockIdx.x + threadIdx.x;
int imsize = m*n;
int loop_limit = D*m*n;

while(col < n)
{
int row = blockDim.y * blockIdx.y + threadIdx.y;
while(row < m)
{
int min_ind = -1;
float current_min = 100000000.0;
int current_val = row * n + col;
int v = 0;

for (int depth = 0; depth < loop_limit; depth+=imsize){

if (dp[depth + current_val] < current_min)
{
min_ind = v;
current_min = dp[depth + current_val];
}
v++;
}
stereo_im[current_val] = min_ind;
row+=blockDim.y;
}
col+=blockDim.x;
}
}