#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2



// dp - cost aggregation array
// cost_image - m x n x D array
// d - use every d channels of input to conserve register memory
// m - image rows
// n - image columns
// D - depth
// depth_stride - pitch along depth dimension
// row_stride - pitch along row dimension


__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float * d_zero, float * d_one, float * d_two, float * d_three){
*d_zero = dp[ind];
if (d > 0)
*d_one = dp[ind - depth_dim_size] + P_one;
else
*d_one = 10000000;

if (d < D-1)
*d_two = dp[ind + depth_dim_size] + P_one;
else
*d_two = 10000000;
return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;

}
__global__ void __r_aggregate(float *dp, float *cost_image, int m, int n)
{
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x;
int depth_dim_size = m*n;
__shared__ float MinArray[SHMEM_SIZE][SHMEM_SIZE];
int K = 0; // this variable keeps track of the progress in aggregating
// across the columns of the image

while ((col < n) & (row < m))
{
int ind = row * n + col;
float prev_min = 100000000.0;

for (int depth = 0; depth < D; depth+=D_STEP){
prev_min = fminf(dp[ind], prev_min);
ind += (depth_dim_size * D_STEP);
}

MinArray[threadIdx.y][threadIdx.x] = prev_min;
__syncthreads();

float d0 = 0;
float d1 = 0;
float d2 = 0;

// when processing a video stream, need to make sure that processing of multiple
// frames can overlap, since after this point only one warp of threads is executing

// threads from only one warp will handle rightward aggregation across the
// region that has been loaded into shared memory
// for threads where threadIdx.y is 0, now threadIdx.x will index the rows
if (threadIdx.y == 0)
{
int agg_row = threadIdx.x + blockIdx.y * blockDim.y;
int start_K = K;
int local_K = 0;

if (agg_row < m)
{
for(; (K < (n - 1)) && (K < (start_K + SHMEM_SIZE)); K++)
{
float d3 = MinArray[threadIdx.x][local_K] + (float) P2;

int ind = agg_row * n + K + 1;
for (int d = 0; d < D; d+=D_STEP){
dp[ind] += cost_image[ind] + dp_criteria(dp, ind-1, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
//dp[ind] = cost_image[ind] + dp[ind - 1];
ind += (depth_dim_size * D_STEP);
}
local_K++;
}
}
}

__syncthreads();
col+=blockDim.x;

}

}