#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2



// dp - cost aggregation array
// cost_image - m x n x D array
// d - use every d channels of input to conserve register memory
// m - image rows
// n - image columns
// D - depth
// depth_stride - pitch along depth dimension
// row_stride - pitch along row dimension


__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float * d_zero, float * d_one, float * d_two, float * d_three){
*d_zero = dp[ind];
if (d > 0)
*d_one = dp[ind - depth_dim_size] + P_one;
else
*d_one = 10000000;

if (d < D-1)
*d_two = dp[ind + depth_dim_size] + P_one;
else
*d_two = 10000000;
return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;

}
__global__ void __diagonal_tl_br_aggregate(float *dp, float *cost_image, int m, int n)
{
// which column of array to work on
int start_col = blockDim.x * blockIdx.x + threadIdx.x + 1;
int depth_dim_size = m*n;

// todo: maybe it will work better to take running average of every d
// slices
while(start_col < n)
{
int col = start_col;
for (int row = 1; row < m; row++)
{
//int arr_ind = 0;
float prev_min = 100000000.0;
int ind = (row - 1) * n + col - 1;

// calculate min cost disparity for this column from row-1
//#pragma unroll
for (int depth = 0; depth < D; depth+=D_STEP){
prev_min = fminf(dp[ind], prev_min);
ind += (depth_dim_size * D_STEP);
}

float d0 = 0;
float d1 = 0;
float d2 = 0;
float d3 = prev_min + (float) P2;
ind = (row - 1) * n + col - 1;
int current_ind = row * n + col;


// todo: try having this loop go from 1 to d-1 and removing the if else
for (int d = 0; d < D; d+=D_STEP){
// for each d I need dp[{d-1, d, d+1}, row-1, col],
dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
ind += (depth_dim_size * D_STEP);
current_ind += (depth_dim_size * D_STEP);
}

col += 1;
if (col == n) // wrap each thread around once it gets to the last column
col = 1;

}
start_col += blockDim.x;
}
}