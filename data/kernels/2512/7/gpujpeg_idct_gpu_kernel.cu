#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void gpujpeg_idct_gpu_kernel_inplace(float* V8)
{
//costants which are used more than once
const float koeficient[6] = {0.4142135623f, 0.3535533905f, 0.4619397662f, 0.1989123673f, 0.7071067811f, -2.0f};

V8[2] *= 0.5411961f;
V8[4] *= 0.509795579f;
V8[5] *= 0.601344887f;

V8[1] = (V8[0] - V8[1]) * koeficient[1];
V8[0] = V8[0] * koeficient[4] - V8[1];

V8[3] = V8[2] * koeficient[1] + V8[3] * koeficient[2];
V8[2] = V8[3] * koeficient[0] - V8[2];

V8[6] = V8[5] * koeficient[2] + V8[6] * koeficient[0];
V8[5] = -0.6681786379f * V8[6] + V8[5];

V8[7] = V8[4] * koeficient[3] + V8[7] * 0.49039264f;
V8[4] = V8[7] * koeficient[3] - V8[4];

//instead of float tmp = V8[1]; V8[1] = V8[2] + V8[1]; V8[2] = tmp - V8[2];
//we use this two operations (with a use of a multiply-add instruction)
V8[1] = V8[2] + V8[1];
V8[2] = koeficient[5] * V8[2] + V8[1];

V8[4] = V8[5] + V8[4];
V8[5] = 2.0f * V8[5] - V8[4];

V8[7] = V8[6] + V8[7];
V8[6] = koeficient[5] * V8[6] + V8[7];

V8[0] = V8[3] + V8[0];
V8[3] = koeficient[5] * V8[3] + V8[0];

V8[5] = V8[6] * koeficient[0] + V8[5];
V8[6] = V8[5] * -koeficient[4] + V8[6];
V8[5] = V8[6] * koeficient[0] + V8[5];

V8[3] = V8[3] + V8[4];
V8[4] = koeficient[5] * V8[4] + V8[3];

V8[2] = V8[2] + V8[5];
V8[5] = koeficient[5] * V8[5] + V8[2];

V8[1] = V8[6] + V8[1];
V8[6] = koeficient[5] * V8[6] + V8[1];

V8[0] = V8[0] + V8[7];
V8[7] = koeficient[5] * V8[7] + V8[0];
}
__global__ void gpujpeg_idct_gpu_kernel(int16_t* source, uint8_t* result, int output_stride, uint16_t* quantization_table)
{
//here the grid is assumed to be only in x - it saves a few operations; if a larger
//block count is used (e. g. GPUJPEG_IDCT_BLOCK_Z == 1), it would need to be adjusted,
//the blockIdx.x not to exceed 65535. In the current state this function is good
//enough for a 67.1 MPix picture (8K is 33.1 MPix)

//the first block of picture processed in this thread block
unsigned int picBlockNumber = (blockIdx.x) * GPUJPEG_IDCT_BLOCK_Y * GPUJPEG_IDCT_BLOCK_X
* GPUJPEG_IDCT_BLOCK_Z;

//pointer to the begin of data for this thread block
int16_t* sourcePtr = (int16_t*) (source) + picBlockNumber * 8;

__shared__ float data[GPUJPEG_IDCT_BLOCK_Z][8][GPUJPEG_IDCT_BLOCK_Y][GPUJPEG_IDCT_BLOCK_X + 1];

//variables to be used later more times (only one multiplication here)
unsigned int z64 = threadIdx.z * 64;
unsigned int x8 = threadIdx.x * 8;

//data copying global -> shared, type casting int16_t -> float and dequantization.
//16b reading gives only 50% efectivity but another ways are too complicated
//so this proves to be the fastest way
#pragma unroll
for (int i = 0; i < 8; i++) {
data[threadIdx.z][i][threadIdx.x][threadIdx.y] = sourcePtr[x8
+ threadIdx.y + i * GPUJPEG_IDCT_BLOCK_X * GPUJPEG_IDCT_BLOCK_Y + z64 * 8]
* quantization_table[threadIdx.x * 8 + threadIdx.y];
}

__syncthreads();

float x[8];

//kompilator delal hrozne psi kusy - zbytecne kopirovani konstant do
//registru atp., bylo jednodussi napsat to v assembleru nez snazit se ho
//presvedcit, aby nedelal blbosti; vsechny konstanty se pouzivaji primo
//hodnotou, nestrkaji se zbytecne do registru

//here the data are being processed by columns - each thread processes one column
#if GPUJPEG_IDCT_USE_ASM
GPUJPEG_IDCT_GPU_KERNEL_INPLACE(data[threadIdx.z][threadIdx.x][0][threadIdx.y],
data[threadIdx.z][threadIdx.x][4][threadIdx.y],
data[threadIdx.z][threadIdx.x][6][threadIdx.y],
data[threadIdx.z][threadIdx.x][2][threadIdx.y],
data[threadIdx.z][threadIdx.x][7][threadIdx.y],
data[threadIdx.z][threadIdx.x][5][threadIdx.y],
data[threadIdx.z][threadIdx.x][3][threadIdx.y],
data[threadIdx.z][threadIdx.x][1][threadIdx.y],

data[threadIdx.z][threadIdx.x][0][threadIdx.y],
data[threadIdx.z][threadIdx.x][1][threadIdx.y],
data[threadIdx.z][threadIdx.x][2][threadIdx.y],
data[threadIdx.z][threadIdx.x][3][threadIdx.y],
data[threadIdx.z][threadIdx.x][4][threadIdx.y],
data[threadIdx.z][threadIdx.x][5][threadIdx.y],
data[threadIdx.z][threadIdx.x][6][threadIdx.y],
data[threadIdx.z][threadIdx.x][7][threadIdx.y])
#else
x[0] = data[threadIdx.z][threadIdx.x][0][threadIdx.y];
x[1] = data[threadIdx.z][threadIdx.x][4][threadIdx.y];
x[2] = data[threadIdx.z][threadIdx.x][6][threadIdx.y];
x[3] = data[threadIdx.z][threadIdx.x][2][threadIdx.y];
x[4] = data[threadIdx.z][threadIdx.x][7][threadIdx.y];
x[5] = data[threadIdx.z][threadIdx.x][5][threadIdx.y];
x[6] = data[threadIdx.z][threadIdx.x][3][threadIdx.y];
x[7] = data[threadIdx.z][threadIdx.x][1][threadIdx.y];

gpujpeg_idct_gpu_kernel_inplace(x);

data[threadIdx.z][threadIdx.x][0][threadIdx.y] = x[0];
data[threadIdx.z][threadIdx.x][1][threadIdx.y] = x[1];
data[threadIdx.z][threadIdx.x][2][threadIdx.y] = x[2];
data[threadIdx.z][threadIdx.x][3][threadIdx.y] = x[3];
data[threadIdx.z][threadIdx.x][4][threadIdx.y] = x[4];
data[threadIdx.z][threadIdx.x][5][threadIdx.y] = x[5];
data[threadIdx.z][threadIdx.x][6][threadIdx.y] = x[6];
data[threadIdx.z][threadIdx.x][7][threadIdx.y] = x[7];
#endif
//between data writing and sync it's good to compute something useful
// - the sync will be shorter.

//output pointer (the begin for this thread block)
unsigned int firstByteOfActualBlock = x8 + z64 + picBlockNumber;

//output pointer for this thread + output row shift; each thread writes 1 row of an
//output block (8B), threads [0 - 7] in threadIdx.x write blocks next to each other,
//threads [1 - 7] in threadIdx.y write next rows of a block; threads [0 - 1] in
//threadIdx.z write next 8 blocks
uint8_t* resultPtr = ((uint8_t*) result) + firstByteOfActualBlock
+ (threadIdx.y + ((firstByteOfActualBlock / output_stride) * 7))
* output_stride;

__syncthreads();

#if GPUJPEG_IDCT_USE_ASM
//here the data are being processed by rows - each thread processes one row
GPUJPEG_IDCT_GPU_KERNEL_INPLACE(data[threadIdx.z][threadIdx.x][threadIdx.y][0],
data[threadIdx.z][threadIdx.x][threadIdx.y][4],
data[threadIdx.z][threadIdx.x][threadIdx.y][6],
data[threadIdx.z][threadIdx.x][threadIdx.y][2],
data[threadIdx.z][threadIdx.x][threadIdx.y][7],
data[threadIdx.z][threadIdx.x][threadIdx.y][5],
data[threadIdx.z][threadIdx.x][threadIdx.y][3],
data[threadIdx.z][threadIdx.x][threadIdx.y][1],

x[0], x[1], x[2], x[3], x[4], x[5], x[6], x[7])
#else
x[0] = data[threadIdx.z][threadIdx.x][threadIdx.y][0];
x[1] = data[threadIdx.z][threadIdx.x][threadIdx.y][4];
x[2] = data[threadIdx.z][threadIdx.x][threadIdx.y][6];
x[3] = data[threadIdx.z][threadIdx.x][threadIdx.y][2];
x[4] = data[threadIdx.z][threadIdx.x][threadIdx.y][7];
x[5] = data[threadIdx.z][threadIdx.x][threadIdx.y][5];
x[6] = data[threadIdx.z][threadIdx.x][threadIdx.y][3];
x[7] = data[threadIdx.z][threadIdx.x][threadIdx.y][1];

gpujpeg_idct_gpu_kernel_inplace(x);
#endif

//output will be written by 8B (one row) which is the most effective way
uint64_t tempResult;
uint64_t* tempResultP = &tempResult;

#pragma unroll
for (int i = 0; i < 8; i++) {
//this would be faster but will work only for 100% quality otherwise some values overflow 255
//((uint8_t*) tempResultP)[i] = __float2uint_rz(x[i] + ((float) 128.0));

//cast float to uint8_t with saturation (.sat) which cuts values higher than
//255 to 255 and smaller than 0 to 0; cuda can't use a reg smaller than 32b
//(though it can convert to 8b for the saturation purposes and save to 32b reg)
uint32_t save;
asm("cvt.rni.u8.f32.sat	%0, %1;" : "=r"(save) : "f"(x[i] + ((float) 128.0)));
((uint8_t*) tempResultP)[i] = save;
}

//writing result - one row of a picture block by a thread
*((uint64_t*) resultPtr) = tempResult;
}