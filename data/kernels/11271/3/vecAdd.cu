#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
c[id] = a[id] + b[id];
}