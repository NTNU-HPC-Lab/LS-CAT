#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
int i = blockIdx.x*blockDim.x+threadIdx.x;

//for (i = 0; i < n; ++i)
if (i < n)
c[i] = a[i] + b[i];
}