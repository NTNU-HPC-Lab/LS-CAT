#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA Kernel function to add the elements of two arrays on the GPU

__global__ void add(int n, float *x, float *y)
{

int index = threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < n; i+= stride)
y[i] = x[i] + y[i];
}