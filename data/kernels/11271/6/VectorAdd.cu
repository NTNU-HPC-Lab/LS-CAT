#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
// Get our global thread ID
int i = blockIdx.x*blockDim.x+threadIdx.x;

//for (i = 0; i < n; ++i) // replaced

// Make sure we do not go out of bounds
if (i < n)
c[i] = a[i] + b[i];
}