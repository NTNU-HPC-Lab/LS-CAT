#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, float *x, float *y)
{

int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n)
y[i] = x[i] + y[i];
}