#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 1000000

using namespace std;


__global__ void RoyFloyd(int* matrix, int k, int N)
{
int i = blockDim.y * blockIdx.y + threadIdx.y;
int j = blockDim.x * blockIdx.x + threadIdx.x;

if (matrix[i*N + k] + matrix[k*N + j] < matrix[i*N + j])
matrix[i*N + j] = matrix[i*N + k] + matrix[k*N + j];
}