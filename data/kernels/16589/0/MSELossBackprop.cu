#include "hip/hip_runtime.h"
#include "includes.h"





using namespace std;

/*** Definitions ***/
// Block width for CUDA kernels
#define BW 128
#define RANDOM_SEED -1

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

__global__ void MSELossBackprop(float *grad_data, float *output, float *target, float *mask, int batch_size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= batch_size)
return;

// const int label_value = static_cast<int>(label[idx]);

// For each item in the batch, decrease the result of the label's value by 1
// diff[idx * num_labels + label_value] -= 1.0f;
if(mask[idx] == -1.0)
grad_data[idx] =  0.05 * (output[idx] - target[idx]);
else if(mask[idx] == 1.0)
grad_data[idx] = 5.0 * (output[idx] - target[idx]);
else
grad_data[idx] = 0.0;
}