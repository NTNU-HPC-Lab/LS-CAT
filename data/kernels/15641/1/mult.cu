#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult(int* A,int* B,int* C) {
int x = threadIdx.x;
int y = threadIdx.y;

if ( x >= N || y >= M )
return;

for(int i=0,j=0; i < N && j < M ; i++, j++) {
C[x*N+y] += A[x*N+j]*B[i*N+y];
}
}