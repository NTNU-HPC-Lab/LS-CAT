#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v3(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int BX = blockDim.x; //same as THEAD_MAX
int i  = bx*(BX*2)+tx;

__shared__ float S[THEAD_MAX];

S[tx] = in[i] + in[i+BX]; //Increased part thread activity at start and start only half the threads
__syncthreads();
for(int s=BX/2; s>0 ;s>>=1){
if(tx < s)
S[tx] += S[tx+s];
__syncthreads();
}
if(tx==0)
out[bx] = S[0];
}