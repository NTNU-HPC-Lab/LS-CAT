#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void warp_reduce(float* S,int tx){
S[tx] += S[tx + 32]; __syncthreads();
S[tx] += S[tx + 16]; __syncthreads();
S[tx] += S[tx + 8];  __syncthreads();
S[tx] += S[tx + 4];  __syncthreads();
S[tx] += S[tx + 2];  __syncthreads();
S[tx] += S[tx + 1];  __syncthreads();
}
__global__ void reduce_v4(float* in,float* out, int n){
int tx = threadIdx.x;
int bx = blockIdx.x;
int BX = blockDim.x; //same as THEAD_MAX
int i  = bx*(BX*2)+tx;

__shared__ float S[THEAD_MAX];

S[tx] = in[i] + in[i+BX]; //Increased part thread activity at start and start only half the threads
__syncthreads();
for(int s=BX/2; s>WARP_SIZE ;s>>=1){
if(tx < s)
S[tx] += S[tx+s];
__syncthreads();
}
if(tx < WARP_SIZE)
warp_reduce(S,tx);				//Unroaling the last warp
if(tx==0)
out[bx] = S[0];
}