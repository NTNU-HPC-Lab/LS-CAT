#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v2(float* a,float* b, int n){

int tx = threadIdx.x;
int ty = threadIdx.y;

int bx = blockIdx.x;
int by = blockIdx.y;

int i = bx*BX + tx;
int j = by*BY + ty;

__shared__ float tile[BY][BX];

if(i >= n || j >= n) return;

tile[ty][tx] = a[j*n+i];

__syncthreads();

i = by*BY + tx;
j = bx*BX + ty;

b[j*n+i] = tile[tx][ty];

}