#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* arr,int offset_min,int n){

int bx = blockIdx.x;
int tx = threadIdx.x;

int BX = blockDim.x;

int i = bx*BX+tx;

if (i>= n|| i < 0) return;
//printf("%d %d - %d %d\n",offset_min,offset_max,i+offset_min,i);
arr[i+offset_min] += 1;

}