#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void offset_access(float* a,int s,int max){
int idx = blockIdx.x*blockDim.x + threadIdx.x;
if (idx+s >= max) return;
a[idx+s] = a[idx+s] + 1;
}