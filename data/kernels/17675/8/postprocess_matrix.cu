#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void postprocess_matrix(float* matrix, long* long_indices, int* indices, unsigned int N_POINTS, unsigned int K)
{
register int TID = threadIdx.x + blockIdx.x * blockDim.x;
if (TID >= N_POINTS*K) return;

// Set pij to 0 for each of the broken values - Note: this should be handled in the ComputePijKernel now
// if (matrix[TID] == 1.0f) matrix[TID] = 0.0f;
indices[TID] = (int) long_indices[TID];
return;
}