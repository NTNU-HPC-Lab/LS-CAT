#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setToOnes(float *data, int size)
{
int index = threadIdx.x + blockIdx.x * blockDim.x; // 1D grid of 1D blocks
if (index < size) data[index] = 1;
}