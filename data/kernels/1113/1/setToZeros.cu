#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setToZeros(float *data, int size)
{
int index = threadIdx.x + blockIdx.x * blockDim.x; // 1D grid of 1D blocks
if (index < size) data[index] = 0;
}