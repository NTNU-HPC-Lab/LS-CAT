#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_1D_3D()
{
return blockIdx.x * blockDim.x * blockDim.y * blockDim.z + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void kernel_1D_3D()
{
printf("Local thread IDs: (%i,%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, threadIdx.z, getGlobalIdx_1D_3D());
}