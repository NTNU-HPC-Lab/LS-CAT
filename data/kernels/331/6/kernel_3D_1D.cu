#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_3D_1D()
{
int blockId = blockIdx.x
+ blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;

int threadId = blockId * blockDim.x + threadIdx.x;

return threadId;
}
__global__ void kernel_3D_1D()
{
printf("Local thread ID: %i   Global thread ID: %i\n", threadIdx.x, getGlobalIdx_3D_1D());
}