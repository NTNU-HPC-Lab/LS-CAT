#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_3D_3D()
{
int blockId = blockIdx.x
+ blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;

int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.z * (blockDim.x * blockDim.y))
+ (threadIdx.y * blockDim.x)
+ threadIdx.x;

return threadId;
}
__global__ void kernel_3D_3D()
{
printf("Local thread IDs: (%i,%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, threadIdx.z, getGlobalIdx_3D_3D());
}