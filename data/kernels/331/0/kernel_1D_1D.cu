#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_1D_1D()
{
return blockIdx.x *blockDim.x + threadIdx.x;
}
__global__ void kernel_1D_1D()
{
printf("Local thread ID: %i   Global thread ID: %i\n", threadIdx.x, getGlobalIdx_1D_1D());
}