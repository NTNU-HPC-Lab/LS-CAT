#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_3D_2D()
{
int blockId = blockIdx.x
+ blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;

int threadId = blockId * (blockDim.x * blockDim.y)
+ (threadIdx.y * blockDim.x)
+ threadIdx.x;

return threadId;
}
__global__ void kernel_3D_2D()
{
printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, getGlobalIdx_3D_2D());
}