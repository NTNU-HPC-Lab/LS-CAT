#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int getGlobalIdx_2D_2D()
{
int blockId = blockIdx.x
+ blockIdx.y * gridDim.x;

int threadId = blockId * (blockDim.x * blockDim.y)
+ (threadIdx.y * blockDim.x)
+ threadIdx.x;

return threadId;
}
__global__ void CudaTransform(unsigned char* dev_img, unsigned int *dev_accu, int w, int h){


//calculate index which this thread have to process
unsigned int index = getGlobalIdx_2D_2D();

//check index is in image bounds
if(index < (w*h)){
//calculate params
float hough_h = ((sqrt(2.0) * (float)(h>w?h:w)) / 2.0);

float center_x = w/2;
float center_y = h/2;

//calculate coordinates for corresponding index in entire image
int x = index % w;
int y = index / w;

if( dev_img[index] > 250 ){ //se il punto è bianco (val in scala di grigio > 250)
for(int t=0;t<180;t++){ //plot dello spazio dei parametri da 0° a 180° (sist. polare)

float r = ( ((float)x - center_x) * cos((float)t * DEG2RAD)) + (((float)y - center_y) * sin((float)t * DEG2RAD));

//dev_accu[ (int)((round(r + hough_h) * 180.0)) + t]++;
atomicAdd(&(dev_accu[ (int)((round(r + hough_h) * 180.0)) + t]), 1);

}
}
}

}