#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_2D_2D()
{
int blockId = blockIdx.x
+ blockIdx.y * gridDim.x;

int threadId = blockId * (blockDim.x * blockDim.y)
+ (threadIdx.y * blockDim.x)
+ threadIdx.x;

return threadId;
}
__global__ void kernel_2D_2D()
{
printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, getGlobalIdx_2D_2D());
}