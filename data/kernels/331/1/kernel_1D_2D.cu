#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_1D_2D()
{
return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void kernel_1D_2D()
{
printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, getGlobalIdx_1D_2D());
}