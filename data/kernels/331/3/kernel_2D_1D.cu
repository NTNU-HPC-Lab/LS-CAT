#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_2D_1D()
{
int blockId = blockIdx.y * gridDim.x
+ blockIdx.x;

int threadId = blockId * blockDim.x + threadIdx.x;

return threadId;
}
__global__ void kernel_2D_1D()
{
printf("Local thread ID: %i   Global thread ID: %i\n", threadIdx.x, getGlobalIdx_2D_1D());
}