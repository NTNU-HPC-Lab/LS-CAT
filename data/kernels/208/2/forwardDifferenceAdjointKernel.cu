#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifferenceAdjointKernel(const int len, const float* source, float* target) {
for (int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1;
idx += blockDim.x * gridDim.x) {
target[idx] = -source[idx] + source[idx - 1];
}
}