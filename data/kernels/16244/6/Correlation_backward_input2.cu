#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Correlation_backward_input2(int item, float *gradInput2, int nInputChannels, int inputHeight, int inputWidth, float *gradOutput, int nOutputChannels, int outputHeight, int outputWidth, float *rInput1, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
// n (batch size), c (num of channels), y (height), x (width)

int n = item;
int y = blockIdx.x * stride1 + pad_size;
int x = blockIdx.y * stride1 + pad_size;
int c = blockIdx.z;

int tch_off = threadIdx.x;

int kernel_rad = (kernel_size - 1) / 2;
int displacement_rad = max_displacement / stride2;
int displacement_size = 2 * displacement_rad + 1;

int pInputWidth = inputWidth + 2 * pad_size;
int pInputHeight = inputHeight + 2 * pad_size;

int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
int pdimxc = pInputWidth * nInputChannels;
int pdimc = nInputChannels;

int tdimcyx = nOutputChannels * outputHeight * outputWidth;
int tdimyx = outputHeight * outputWidth;
int tdimx = outputWidth;

int odimcyx = nInputChannels * inputHeight* inputWidth;
int odimyx = inputHeight * inputWidth;
int odimx = inputWidth;

float nelems = kernel_size * kernel_size * nInputChannels;

__shared__ float prod_sum[CUDA_NUM_THREADS];
prod_sum[tch_off] = 0;

for (int tc = tch_off; tc < nOutputChannels; tc += CUDA_NUM_THREADS) {
int i2 = (tc % displacement_size - displacement_rad) * stride2;
int j2 = (tc / displacement_size - displacement_rad) * stride2;

int xmin = (x - kernel_rad - max_displacement - i2) / stride1;
int ymin = (y - kernel_rad - max_displacement - j2) / stride1;

int xmax = (x + kernel_rad - max_displacement - i2) / stride1;
int ymax = (y + kernel_rad - max_displacement - j2) / stride1;

if (xmax < 0 || ymax < 0 || xmin >= outputWidth || ymin >= outputHeight) {
// assumes gradInput2 is pre-allocated and zero filled
continue;
}

if (xmin > xmax || ymin > ymax) {
// assumes gradInput2 is pre-allocated and zero filled
continue;
}

xmin = max(0,xmin);
xmax = min(outputWidth-1,xmax);

ymin = max(0,ymin);
ymax = min(outputHeight-1,ymax);

int indx1 = n * pdimyxc + (y - j2)* pdimxc + (x - i2) * pdimc + c;
float val1 = rInput1[indx1];

for (int j = ymin; j <= ymax; ++j) {
for (int i = xmin; i <= xmax; ++i) {
int tindx = n * tdimcyx + tc * tdimyx + j * tdimx + i;
prod_sum[tch_off] += gradOutput[tindx] * val1;
}
}
}

__syncthreads();

if(tch_off == 0) {
float reduce_sum = 0;
for(int idx = 0; idx < CUDA_NUM_THREADS; idx++) {
reduce_sum += prod_sum[idx];
}
const int indx2 = n * odimcyx + c * odimyx + (y - pad_size) * odimx + (x - pad_size);
gradInput2[indx2] = reduce_sum / nelems;
}

}