#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
int j = threadIdx.x + blockDim.x * blockIdx.x;
int i = threadIdx.y + blockDim.y * blockIdx.y;

if (i >= h || j >= w) return;

int flow_row_offset = i * flow_stride;
int image_row_offset = i * image_stride;

float u_ = u[flow_row_offset + j];
float v_ = v[flow_row_offset + j];

//bottom left corner of target pixel
float cx = u_ * time_scale + (float)j + 1.0f;
float cy = v_ * time_scale + (float)i + 1.0f;
// pixel containing bottom left corner
int tx = __float2int_rn (cx);
int ty = __float2int_rn (cy);

float value = src[image_row_offset + j];
// fill pixel
if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
{
atomicAdd (dst + ty * image_stride + tx, value);
}
}