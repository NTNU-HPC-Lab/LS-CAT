#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createLookupKernel(const int* inds, int total, int* output)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if (idx < total)
output[inds[idx]] = idx;
}