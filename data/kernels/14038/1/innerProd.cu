#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void innerProd(float *aa, float *bb, float *cc)
{
__shared__ float temp[THREADS_PER_BLOCK];
int index = threadIdx.x + blockIdx.x* blockDim.x;
temp[threadIdx.x] = aa[index]*bb[index];

*cc = 0; // Initialized to avoid memory problems. See comments
// below, next to the free and hipFree commands.

// No thread goes beyond this point until all of them
// have reached it. Threads are only synchronized within
// a block.
__syncthreads();

//  Thread 0 sums the pairwise products
if (threadIdx.x == 0) {
float sum = 0;
for (int i = 0; i < THREADS_PER_BLOCK; i++){
sum += temp[i];
}
// Use atomicAdd to avoid different blocks accessing cc at the
// same time (race condition). The atomic opperation enables
// read-modify-write to be performed by a block without interruption.
//*cc += sum;
atomicAdd(cc, sum);
}

}