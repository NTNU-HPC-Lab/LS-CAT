#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef _KERNEL_H
#define _KERNEL_H
typedef struct Node {
int starting;
int no_of_edges;
}Node;



#endif
__global__ void test(Node* d_graph_nodes, int no_of_nodes) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < no_of_nodes) {
d_graph_nodes[tid].starting+=1;
}
}