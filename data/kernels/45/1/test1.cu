#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef _KERNEL_H
#define _KERNEL_H
typedef struct Node {
int starting;
int no_of_edges;
}Node;



#endif
__global__ void test1(bool* d_graph_visited, int no_of_nodes) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < no_of_nodes) {
d_graph_visited[tid] = true;
}
}