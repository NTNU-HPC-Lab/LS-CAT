#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bin_start(int *binStart, int *binEnd, int *partBin, int nparts)
{
// This kernel function was adapted from NVIDIA CUDA 5.5 Examples
// This software contains source code provided by NVIDIA Corporation
extern __shared__ int sharedBin[];    //blockSize + 1
int index = threadIdx.x + blockIdx.x*blockDim.x;
int bin;

// for a given bin index, the previous bins's index is stored in sharedBin
if (index < nparts) {
bin = partBin[index];

// Load bin data into shared memory so that we can look
// at neighboring particle's hash value without loading
// two bin values per thread
sharedBin[threadIdx.x + 1] = bin;

if (index > 0 && threadIdx.x == 0) {
// first thread in block must load neighbor particle bin
sharedBin[0] = partBin[index - 1];
}
}
__syncthreads();

if (index < nparts) {
// If this particle has a different cell index to the previous
// particle then it must be the first particle in the cell,
// so store the index of this particle in the cell.
// As it isn't the first particle, it must also be the cell end of
// the previous particle's cell
bin = partBin[index];

if (index == 0 || bin != sharedBin[threadIdx.x]) {
binStart[bin] = index;

if (index > 0)
binEnd[sharedBin[threadIdx.x]] = index;
}

if (index == nparts - 1)
{
binEnd[bin] = index + 1;
}
}
}