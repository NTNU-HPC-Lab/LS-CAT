#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hello2D()
{
int blocksize = blockIdx.y * blockIdx.x;
int blockId = gridDim.x * blockIdx.y + blockIdx.x;
int tid = blockId * blocksize + blockDim.x * threadIdx.y + threadIdx.x;

printf("I am thread (%d, %d) in block (%d, %d). Global thread ID = %d\n", threadIdx.y, threadIdx.x, blockIdx.y, blockIdx.x, tid);
}