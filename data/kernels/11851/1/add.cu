#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *c, float* a, float *b, int values){
int blockD = blockDim.x;
int blockX = blockIdx.x;
int threadX = threadIdx.x;

int i = blockX * blockD + threadX;
if(i < values)
c[i] = a[i] + b[i];
//printf("Hello Im thread %d in block %d of %d threads\n", threadX, blockX, blockD);
}