#include "hip/hip_runtime.h"
#include "includes.h"

// Save wave state as binary file.
__global__ void fillSpaceTSteps(int N, int T, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2)
{

int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

waveSpace[N * i + j] = 2 * waveSpaceTMin1[N * i + j] - waveSpaceTMin2[N * i + j] + (c * c) * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);

__syncthreads();
}