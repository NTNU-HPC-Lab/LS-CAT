#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
////////////////
// TO-DO #5.2 ////////////////////////////////////////////////
// Implement the GPU version of cpu_applyFilter()           //
//                                                          //
// Does it make sense to have a separate gpu_applyFilter()? //
//////////////////////////////////////////////////////////////

return 0.0f;
}
__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{
float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };

int index_x = blockIdx.x * blockDim.x + threadIdx.x;
int index_y = blockIdx.y * blockDim.y + threadIdx.y;

if (index_x < (width - 2) && index_y < (height - 2))
{
int offset_t = index_y * width + index_x;
int offset   = (index_y + 1) * width + (index_x + 1);

image_out[offset] = gpu_applyFilter(&image[offset_t],
width, gaussian, 3);
}
}