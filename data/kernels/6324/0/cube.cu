#include "hip/hip_runtime.h"
#include "includes.h"

// this is how cuda knows that this code is a kernel by calling __global__

__global__ void cube(float * d_out, float * d_in) {
int idx = threadIdx.x ;
float f = d_in[idx];
d_out[idx] = f * f * f;
}