#include "hip/hip_runtime.h"
#include "includes.h"
/*
Sample input file format:

1.Line : 6 => Number of nodes(int)
2.Line : 7 => Number of edges(int)
3.Line : 1 2 5.0 ----------------
4.Line : 2 3 1.5                |
5.Line : 1 3 2.1				|
6.Line : 1 4 1.2				|=> Edges
7.Line : 1 5 15.5				|
8.Line : 2 5 3.6				|
9.Line : 3 6 1.2-----------------
10.Line : 1 => Start node.
///////////////////////////////////////////////////////

Doesn't check any error condition.
*/


using namespace std;

// Edge struct.
typedef struct {
int* startPoints;
int* endPoints;
double* weights;
}Edge;


// This kernel will call queue size thread.

__global__ void updateQueueKernel(int *queueu,int *queueSize, const int *startPoints, const int *endPoints,const int*visitedArray, const int *currentVertex ) {

int index = threadIdx.x;
if (startPoints[index] == *currentVertex && visitedArray[endPoints[index]] == 0 )
{
int oldValue = atomicAdd(queueSize,1);
queueu[oldValue] = index;
}
}