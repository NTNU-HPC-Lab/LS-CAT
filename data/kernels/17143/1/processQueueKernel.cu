#include "hip/hip_runtime.h"
#include "includes.h"
/*
Sample input file format:

1.Line : 6 => Number of nodes(int)
2.Line : 7 => Number of edges(int)
3.Line : 1 2 5.0 ----------------
4.Line : 2 3 1.5                |
5.Line : 1 3 2.1				|
6.Line : 1 4 1.2				|=> Edges
7.Line : 1 5 15.5				|
8.Line : 2 5 3.6				|
9.Line : 3 6 1.2-----------------
10.Line : 1 => Start node.
///////////////////////////////////////////////////////

Doesn't check any error condition.
*/


using namespace std;

// Edge struct.
typedef struct {
int* startPoints;
int* endPoints;
double* weights;
}Edge;


// This kernel will call queue size thread.

__global__ void processQueueKernel(int *parentArray, double *resultWeightArray, const int* queue,const int *startPoints,const int *endPoints, const double *weightArray) {

int threadIndex = threadIdx.x;
int elementIndex = queue[threadIndex];
int startNode = startPoints[elementIndex];
int endNode = endPoints[elementIndex];
double edgeWeight = weightArray[elementIndex];
double nodeWeight = resultWeightArray[startNode];

if (nodeWeight + edgeWeight < resultWeightArray[endNode])
{
resultWeightArray[endNode] = nodeWeight + edgeWeight;
parentArray[endNode] = startNode;
}
}