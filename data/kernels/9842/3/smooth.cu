#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
int myIdx = threadIdx.x * gridDim.x + blockIdx.x;
int numThreads = blockDim.x * gridDim.x;
int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
float myElt = v[myIdx];
float myLeftElt = v[myLeftIdx];
float myRightElt = v[myRightIdx];
v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}