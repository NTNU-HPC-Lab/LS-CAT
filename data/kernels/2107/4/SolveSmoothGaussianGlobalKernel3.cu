#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SolveSmoothGaussianGlobalKernel3(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float w[9] = {0.0f, 0.1667f, 0.0f, 0.1667f, 0.3333f, 0.1667f, 0.0f, 0.1667f, 0.0f};

float sumu = 0;
float sumv = 0;
for (int j = 0; j < 3; j++) {
for (int i = 0; i < 3; i++) {
//get values
int col = (ix + i - 1);
int row = (iy + j - 1);
if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
sumu = sumu + w[j * 3 + i] * u[col + stride*row];
sumv = sumv + w[j * 3 + i] * v[col + stride*row];
}
//solve gaussian
}
}
outputu[pos] = sumu;
outputv[pos] = sumv;
outputbku[pos] = bku[pos] + u[pos] - sumu;
outputbkv[pos] = bkv[pos] + v[pos] - sumv;
}