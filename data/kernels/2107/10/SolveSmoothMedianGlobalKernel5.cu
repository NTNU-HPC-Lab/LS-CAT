#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SolveSmoothMedianGlobalKernel5(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float mu[25] = { 0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0 };

float mv[25] = { 0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0,
0, 0, 0, 0, 0 };

for (int j = 0; j < 5; j++) {
for (int i = 0; i < 5; i++) {
//get values
int col = (ix + i - 2);
int row = (iy + j - 2);
if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
mu[j * 5 + i] = u[col + stride*row];
mv[j * 5 + i] = v[col + stride*row];
}
else if ((col < 0) && (row >= 0) && (row < height)) {
mu[j * 5 + i] = u[stride*row];
mv[j * 5 + i] = v[stride*row];
}
else if ((col > width) && (row >= 0) && (row < height)) {
mu[j * 5 + i] = u[width - 1 + stride*row];
mv[j * 5 + i] = v[width - 1 + stride*row];
}
else if ((col >= 0) && (col < width) && (row < 0)) {
mu[j * 5 + i] = u[col];
mv[j * 5 + i] = v[col];
}
else if ((col >= 0) && (col < width) && (row > height)) {
mu[j * 5 + i] = u[col + stride*(height - 1)];
mv[j * 5 + i] = v[col + stride*(height - 1)];
}
//solve gaussian
}
}

float tmpu, tmpv;
for (int j = 0; j < 25; j++) {
for (int i = j+1; i < 25; i++) {
if (mu[j] > mu[i]) {
//Swap the variables.
tmpu = mu[j];
mu[j] = mu[i];
mu[i] = tmpu;
}
if (mv[j] > mv[i]) {
//Swap the variables.
tmpv = mv[j];
mv[j] = mv[i];
mv[i] = tmpv;
}
}
}

outputu[pos] = mu[12];
outputv[pos] = mv[12];
outputbku[pos] = bku[pos] + u[pos] - mu[12];
outputbkv[pos] = bkv[pos] + v[pos] - mv[12];
}