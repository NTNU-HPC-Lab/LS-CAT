#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SolveSmoothMedianGlobalKernel3(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float mu[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

float mv[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

for (int j = 0; j < 3; j++) {
for (int i = 0; i < 3; i++) {
//get values
int col = (ix + i - 1);
int row = (iy + j - 1);
int index = j * 3 + i;
if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
mu[index] = u[col + stride*row];
mv[index] = v[col + stride*row];
}
else if ((col < 0) && (row >= 0) && (row < height)) {
mu[index] = u[stride*row];
mv[index] = v[stride*row];
}
else if ((col > width) && (row >= 0) && (row < height)) {
mu[index] = u[width - 1 + stride*row];
mv[index] = v[width - 1 + stride*row];
}
else if ((col >= 0) && (col < width) && (row < 0)) {
mu[index] = u[col];
mv[index] = v[col];
}
else if ((col >= 0) && (col < width) && (row > height)) {
mu[index] = u[col + stride*(height - 1)];
mv[index] = v[col + stride*(height - 1)];
}
//solve gaussian
}
}

float tmpu, tmpv;
for (int j = 0; j < 9; j++) {
for (int i = j + 1; i < 9; i++) {
if (mu[j] > mu[i]) {
//Swap the variables.
tmpu = mu[j];
mu[j] = mu[i];
mu[i] = tmpu;
}
if (mv[j] > mv[i]) {
//Swap the variables.
tmpv = mv[j];
mv[j] = mv[i];
mv[i] = tmpv;
}
}
}

outputu[pos] = mu[4];
outputv[pos] = mv[4];
outputbku[pos] = bku[pos] + u[pos] - mu[4];
outputbkv[pos] = bkv[pos] + v[pos] - mv[4];
}