#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "SolveSmoothGaussianGlobalKernel5.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *u = NULL;
hipMalloc(&u, XSIZE*YSIZE);
float *v = NULL;
hipMalloc(&v, XSIZE*YSIZE);
float *bku = NULL;
hipMalloc(&bku, XSIZE*YSIZE);
float *bkv = NULL;
hipMalloc(&bkv, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
float *outputu = NULL;
hipMalloc(&outputu, XSIZE*YSIZE);
float *outputv = NULL;
hipMalloc(&outputv, XSIZE*YSIZE);
float *outputbku = NULL;
hipMalloc(&outputbku, XSIZE*YSIZE);
float *outputbkv = NULL;
hipMalloc(&outputbkv, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
SolveSmoothGaussianGlobalKernel5<<<gridBlock,threadBlock>>>(u,v,bku,bkv,width,height,stride,outputu,outputv,outputbku,outputbkv);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
SolveSmoothGaussianGlobalKernel5<<<gridBlock,threadBlock>>>(u,v,bku,bkv,width,height,stride,outputu,outputv,outputbku,outputbkv);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
SolveSmoothGaussianGlobalKernel5<<<gridBlock,threadBlock>>>(u,v,bku,bkv,width,height,stride,outputu,outputv,outputbku,outputbkv);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}