#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SolveSmoothGaussianGlobalKernel5(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float w[25] = { 0.0037, 0.0147, 0.0256, 0.0147, 0.0037,
0.0147, 0.0586, 0.0952, 0.0586, 0.0147,
0.0256, 0.0952, 0.1502, 0.0952, 0.0256,
0.0147, 0.0586, 0.0952, 0.0586, 0.0147,
0.0037, 0.0147, 0.0256, 0.0147, 0.0037 };

float sumu = 0;
float sumv = 0;
for (int j = 0; j < 5; j++) {
for (int i = 0; i < 5; i++) {
//get values
int col = (ix + i - 2);
int row = (iy + j - 2);
if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
sumu = sumu + w[j * 5 + i] * u[col + stride*row];
sumv = sumv + w[j * 5 + i] * v[col + stride*row];
}
//solve gaussian
}
}
outputu[pos] = sumu;
outputv[pos] = sumv;
outputbku[pos] = bku[pos] + u[pos] - sumu;
outputbkv[pos] = bkv[pos] + v[pos] - sumv;
}