#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
uint8_t * outX = out + y * rowSizeOut + x;
(*outX) = ((*in1X) < (*in2X)) ? (*in1X) : (*in2X);
}
}