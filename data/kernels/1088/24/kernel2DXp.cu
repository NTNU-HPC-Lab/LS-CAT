#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2DXp ( double* dataOutput, double* dataInput, const double* weights, const int numSten, const int numStenLeft, const int numStenRight, const int nxLocal, const int nyLocal, const int BLOCK_X, const int nx )
{
// -----------------------------
// Allocate the shared memory
// -----------------------------

extern __shared__ int memory[];

double* arrayLocal = (double*)&memory;
double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

// Move the weigths into shared memory
#pragma unroll
for (int k = 0; k < numSten; k++)
{
weigthsLocal[k] = weights[k];
}

// -----------------------------
// Set the indexing
// -----------------------------

// True matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Local matrix index
int localIdx = threadIdx.x + numStenLeft;
int localIdy = threadIdx.y;

// Local sum variable
double sum = 0.0;

// Set index for summing stencil
int stenSet;

// -----------------------------
// Set interior
// -----------------------------

arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

// -----------------------------
// Set x boundaries
// -----------------------------

// If block is in the interior
if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
{

if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}
}

// If block is on the left boundary
if (blockIdx.x == 0)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (nx - numStenLeft + threadIdx.x)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}
}

// Set the right boundary blocks
if (blockIdx.x == nx / BLOCK_X - 1)
{
arrayLocal[localIdy * nxLocal + threadIdx.x + numStenLeft] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + threadIdx.x];
}
}

// -----------------------------
// Compute the stencil
// -----------------------------

__syncthreads();

stenSet = localIdy * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k];
}

__syncthreads();

// -----------------------------
// Copy back to global
// -----------------------------

dataOutput[globalIdy * nx + globalIdx] = sum;
}