#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setMultiLHS ( double* dsMulti, double* dlMulti, double* diagMulti, double* duMulti, double* dwMulti,  double a, double b, double c, double d, double e,  int nx, int batchCount )
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Index access
int index = globalIdy * batchCount + globalIdx;

if (globalIdx < batchCount && globalIdy < nx)
{

dsMulti[index] = a;

dlMulti[index] = b;

diagMulti[index] = c;

duMulti[index] = d;

dwMulti[index] = e;

}
}