#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void solveFull ( double* data,  double* inv1, double* inv2,  const int nx, const int nBatch )
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Set values to last two entries in array
double oldNx2 = data[(nx - 2) * nBatch + globalIdx]; // Two points from end
double oldNx1 = data[(nx - 1) * nBatch + globalIdx]; // One point from end

// Set index being computed
int index = globalIdy * nBatch + globalIdx;

if (globalIdy < nx - 2)
{
data[index] = data[index] - (inv1[index] * oldNx2 + inv2[index] * oldNx1);
}
}