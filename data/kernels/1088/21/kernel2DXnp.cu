#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2DXnp ( double* dataOutput, double* dataInput, const double* weights, const int numSten, const int numStenLeft, const int numStenRight, const int nxLocal, const int nyLocal, const int BLOCK_X, const int nx )
{
// Allocate the shared memory
extern __shared__ int memory[];

double* arrayLocal = (double*)&memory;
double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

// Move the weigths into shared memory
#pragma unroll
for (int k = 0; k < numSten; k++)
{
weigthsLocal[k] = weights[k];
}

// True matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Local matrix index
int localIdx = threadIdx.x + numStenLeft;
int localIdy = threadIdx.y;

// Local sum variable
double sum = 0.0;

// Set index for summing stencil
int stenSet;

// Set all interior blocks
if (blockIdx.x != 0 && blockIdx.x != nx / (BLOCK_X) - 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}

__syncthreads();


stenSet = localIdy * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k];
}

dataOutput[globalIdy * nx + globalIdx] = sum;
}

// Set all left boundary blocks
if (blockIdx.x == 0)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + threadIdx.x + BLOCK_X] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}

__syncthreads();

if (threadIdx.x >= numStenLeft)
{

stenSet = localIdy * nxLocal + threadIdx.x - numStenLeft;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k];
}

dataOutput[globalIdy * nx + globalIdx] = sum;
}
}

// Set the right boundary blocks
if (blockIdx.x == nx / BLOCK_X - 1)
{
arrayLocal[localIdy * nxLocal + threadIdx.x + numStenLeft] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

__syncthreads();

if (threadIdx.x < BLOCK_X - numStenRight)
{

stenSet = localIdy * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k];
}
}

dataOutput[globalIdy * nx + globalIdx] = sum;
}
}