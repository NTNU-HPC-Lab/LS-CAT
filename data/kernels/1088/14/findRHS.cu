#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void findRHS(double* cOld, double* cCurr, double* cHalf, double* cNonLinRHS, int nx)
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Set index being computed
int index = globalIdy * nx + globalIdx;

// Set the RHS for inversion
cHalf[index] += - (2.0 / 3.0) * (cCurr[index] - cOld[index]) + cNonLinRHS[index];

// Set cOld to cCurr
cOld[index] = cCurr[index];
}