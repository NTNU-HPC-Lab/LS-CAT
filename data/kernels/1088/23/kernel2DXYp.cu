#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2DXYp ( double* dataOutput, double* dataInput, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenHoriz, const int numStenLeft, const int numStenRight, const int numStenVert, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_X, const int BLOCK_Y, const int nx, const int nyTile )
{
// -----------------------------
// Allocate the shared memory
// -----------------------------

extern __shared__ int memory[];

double* arrayLocal = (double*)&memory;
double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

// Move the weigths into shared memory
#pragma unroll
for (int k = 0; k < numSten; k++)
{
weigthsLocal[k] = weights[k];
}

// -----------------------------
// Set the indexing
// -----------------------------

// True matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Local matrix index
int localIdx = threadIdx.x + numStenLeft;
int localIdy = threadIdx.y + numStenTop;

// Local sum variable
double sum = 0.0;

// Set index for summing stencil
int stenSet;

// Set temporary index for looping
int temp;

// Use to loop over indexing in the weighsLocal
int weight = 0;

// -----------------------------
// Set interior
// -----------------------------

arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

// -----------------------------
// Set x boundaries
// -----------------------------

// If block is in the interior
if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
{
if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}
}

// If block is on the left boundary
if (blockIdx.x == 0)
{
if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (nx - numStenLeft + threadIdx.x)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
}
}

// Set the right boundary blocks
if (blockIdx.x == nx / BLOCK_X - 1)
{
if (threadIdx.x < numStenLeft)
{
arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
}

if (threadIdx.x < numStenRight)
{
arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + threadIdx.x];
}
}

// -----------------------------
// Set y boundaries
// -----------------------------

// Set interior y boundary
if (blockIdx.y != 0 && blockIdx.y != nyTile / BLOCK_Y - 1)
{
if (threadIdx.y < numStenTop )
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
}
}

// Set top y boundary
if (blockIdx.y == 0)
{
if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
}
}

// Set bottom y boundary
if (blockIdx.y == nyTile / BLOCK_Y - 1)
{
if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
}
}

// -----------------------------
// Corners - Interior of tile
// -----------------------------

// Set interior y boundary
if (blockIdx.y != 0 && blockIdx.y != nyTile / BLOCK_Y - 1)
{
// If block is in the interior
if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
}
}

// If block is on the left boundary
if (blockIdx.x == 0)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (nx - numStenLeft + threadIdx.x)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (nx - numStenLeft + threadIdx.x)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
}
}

// If block is on the right boundary
if (blockIdx.x == nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + threadIdx.x];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + threadIdx.x];
}
}
}

// -----------------------------
// Corners - Top of tile
// -----------------------------

// Set top y boundary
if (blockIdx.y == 0)
{
// If block is in the interior
if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = boundaryTop[threadIdx.y * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = boundaryTop[threadIdx.y * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
}
}

// If block is on the left boundary
if (blockIdx.x == 0)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = boundaryTop[threadIdx.y * nx + (nx - numStenLeft + threadIdx.x)];

}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = boundaryTop[threadIdx.y * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (nx - numStenLeft + threadIdx.x)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
}
}

// If block is on the right boundary
if (blockIdx.x == nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = boundaryTop[threadIdx.y * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = boundaryTop[threadIdx.y * nx + threadIdx.x];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + threadIdx.x];
}
}
}

// -----------------------------
// Corners - Bottom of tile
// -----------------------------

// Set bottom y boundary
if (blockIdx.y == nyTile / BLOCK_Y - 1)
{
// If block is in the interior
if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = boundaryBottom[threadIdx.y * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] =  boundaryBottom[threadIdx.y * nx + (globalIdx + BLOCK_X)];

}
}

// If block is on the left boundary
if (blockIdx.x == 0)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (nx - numStenLeft + threadIdx.x)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = boundaryBottom[threadIdx.y * nx + (nx - numStenLeft + threadIdx.x)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] =  boundaryBottom[threadIdx.y * nx + (globalIdx + BLOCK_X)];
}
}

// If block is on the right boundary
if (blockIdx.x == nx / BLOCK_X - 1)
{
// Top Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
}

// Top Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + threadIdx.x];
}

// Bottom Left
if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = boundaryBottom[threadIdx.y * nx + (globalIdx - numStenLeft)];
}

// Bottom Right
if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] =  boundaryBottom[threadIdx.y * nx + threadIdx.x];
}
}
}

// -----------------------------
// Compute the stencil
// -----------------------------

__syncthreads();

stenSet = (localIdy - numStenTop) * nxLocal + (localIdx - numStenLeft);
weight = 0;

for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
{
temp = j * nxLocal;

for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
{
sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

weight++;
}
}

__syncthreads();

// -----------------------------
// Copy back to global
// -----------------------------

dataOutput[globalIdy * nx + globalIdx] = sum;

}