#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2DYp ( double* dataOutput, double* dataInput, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_Y, const int nx, const int nyTile )
{
// Allocate the shared memory
extern __shared__ int memory[];

double* arrayLocal = (double*)&memory;
double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

// Move the weigths into shared memory
#pragma unroll
for (int k = 0; k < numSten; k++)
{
weigthsLocal[k] = weights[k];
}

// True matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Local matrix index
int localIdx = threadIdx.x;
int localIdy = threadIdx.y + numStenTop;

// Local sum variable
double sum = 0.0;

// Set index for summing stencil
int stenSet;

// Set all interior blocks
if (blockIdx.y != 0 && blockIdx.y != nyTile / (BLOCK_Y) - 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
}

__syncthreads();


stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataOutput[globalIdy * nx + globalIdx] = sum;
}

// // Set all top boundary blocks
if (blockIdx.y == 0)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
}

__syncthreads();

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataOutput[globalIdy * nx + globalIdx] = sum;
}

// Set the bottom boundary blocks
if (blockIdx.y == nyTile / BLOCK_Y - 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
}

__syncthreads();

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataOutput[globalIdy * nx + globalIdx] = sum;
}
}