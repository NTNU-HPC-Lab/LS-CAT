#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void findNew(double* cCurr, double* cBar, double* cHalf, int nx)
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Set index being computed
int index = globalIdy * nx + globalIdx;

// Recover the new data
cCurr[index] = cBar[index] + cHalf[index];
}