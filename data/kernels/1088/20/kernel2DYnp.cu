#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2DYnp ( double* dataNew, double* dataOld, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_Y, const int nx, const int nyTile, const int tileTop, const int tileBottom )
{
// Allocate the shared memory
extern __shared__ int memory[];

double* arrayLocal = (double*)&memory;
double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

// Move the weigths into shared memory
#pragma unroll
for (int k = 0; k < numSten; k++)
{
weigthsLocal[k] = weights[k];
}

// True matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Local matrix index
int localIdx = threadIdx.x;
int localIdy = threadIdx.y + numStenTop;

// Local sum variable
double sum = 0.0;

// Set index for summing stencil
int stenSet;

// Set all interior blocks
if (blockIdx.y != 0 && blockIdx.y != nyTile / (BLOCK_Y) - 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataOld[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataOld[(globalIdy + BLOCK_Y) * nx + globalIdx];
}

__syncthreads();


stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataNew[globalIdy * nx + globalIdx] = sum;
}

// Set all top boundary blocks
if (blockIdx.y == 0)
{
if (tileTop != 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataOld[(globalIdy + BLOCK_Y) * nx + globalIdx];
}

__syncthreads();

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataNew[globalIdy * nx + globalIdx] = sum;
}
else
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenBottom)
{
arrayLocal[(threadIdx.y + BLOCK_Y) * nxLocal + localIdx] = dataOld[(globalIdy + BLOCK_Y) * nx + globalIdx];
}

__syncthreads();

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

if (threadIdx.y < BLOCK_Y - numStenTop)
{
dataNew[(globalIdy + numStenTop) * nx + globalIdx] = sum;
}
}
}


// Set the bottom boundary blocks
if (blockIdx.y == nyTile / BLOCK_Y - 1)
{
if (tileBottom != 1)
{
arrayLocal[localIdy * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataOld[(globalIdy - numStenTop) * nx + globalIdx];
}

if (threadIdx.y < numStenBottom)
{
arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
}

__syncthreads();

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

dataNew[globalIdy * nx + globalIdx] = sum;
}
else
{
arrayLocal[localIdy * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];

if (threadIdx.y < numStenTop)
{
arrayLocal[threadIdx.y * nxLocal + localIdx] = dataOld[(globalIdy - numStenTop) * nx + globalIdx];
}

stenSet = threadIdx.y * nxLocal + threadIdx.x;

#pragma unroll
for (int k = 0; k < numSten; k++)
{
sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
}

__syncthreads();

if (threadIdx.y < BLOCK_Y - numStenBottom)
{
dataNew[globalIdy * nx + globalIdx] = sum;
}
}
}
}