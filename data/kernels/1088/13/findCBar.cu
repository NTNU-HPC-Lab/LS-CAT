#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void findCBar(double* cOld, double* cCurr, double* cBar, int nx)
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

// Set index being computed
int index = globalIdy * nx + globalIdx;

// Find cBar
cBar[index] = 2.0 * cCurr[index] - cOld[index];
}