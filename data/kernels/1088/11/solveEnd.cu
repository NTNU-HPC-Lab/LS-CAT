#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void solveEnd ( double* data,  const double a, const double b, const double d, const double e,  const double omega_11, const double omega_12, const double omega_21, const double omega_22,  const int nx, const int nBatch )
{
// Matrix index
int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

// Last two vectors
double newNx2;
double newNx1;

// Compute lambda = d^~ - transpose(g) * inverse(E) * d_hat
newNx2 = data[(nx - 2) * nBatch + globalIdx] - (e * data[globalIdx] + a * data[(nx - 4) * nBatch + globalIdx] + b * data[(nx - 3) * nBatch + globalIdx]);
newNx1 = data[(nx - 1) * nBatch + globalIdx] - (d * data[globalIdx] + e * data[nBatch + globalIdx] + a * data[(nx - 3) * nBatch + globalIdx]);

// Compute x^~ = omega * lambda
data[(nx - 2) * nBatch + globalIdx] = omega_11 * newNx2 + omega_12 * newNx1;
data[(nx - 1) * nBatch + globalIdx] = omega_21 * newNx2 + omega_22 * newNx1;
}