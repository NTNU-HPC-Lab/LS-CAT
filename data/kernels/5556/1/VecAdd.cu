#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float* A, const float* B, float* C)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
C[i] = A[i] + B[i];
}