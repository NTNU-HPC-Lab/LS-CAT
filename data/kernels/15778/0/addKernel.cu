#include "hip/hip_runtime.h"
#include "includes.h"





// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}