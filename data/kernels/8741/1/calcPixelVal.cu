#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcPixelVal(float *g_idata, float* constant, float* min)
{
unsigned int i = blockIdx.x * blockDim.x  + threadIdx.x;

if(i<LENA_SIZE)g_idata[i]=(g_idata[i]-(*min))*(*constant);

}