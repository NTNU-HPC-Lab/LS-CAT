#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update(double *weights_in_d, double *weights_h_d, double *weights_out_d, double *weights_in_delta_d, double *weights_h_delta_d, double *weights_out_delta_d, double *error_d){
int tix = threadIdx.x;

if(tix < INPUTS*H_HEIGHT){
weights_in_d[tix] -= (alpha_d * weights_in_delta_d[tix] / 55);
weights_in_delta_d[tix] = 0.0;
}

weights_h_d[tix] -= (alpha_d * weights_h_delta_d[tix] / 55);
weights_h_delta_d[tix] = 0.0;

if(tix < OUTPUTS*H_HEIGHT){
weights_out_d[tix] -= (alpha_d * weights_out_delta_d[tix] / 55);
weights_out_delta_d[tix] = 0.0;
}

if(tix < 1){
error_d[0] = error_d[0] * 100.0 / 55;
printf("\nGPU Error: %f\n", error_d[0]);
error_d[0] = 0;
}

}