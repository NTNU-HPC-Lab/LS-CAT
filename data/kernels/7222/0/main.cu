#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Test.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *training_in_d = NULL;
hipMalloc(&training_in_d, XSIZE*YSIZE);
double *training_out_d = NULL;
hipMalloc(&training_out_d, XSIZE*YSIZE);
double *data_range_d = NULL;
hipMalloc(&data_range_d, XSIZE*YSIZE);
double *weights_in_d = NULL;
hipMalloc(&weights_in_d, XSIZE*YSIZE);
double *weights_out_d = NULL;
hipMalloc(&weights_out_d, XSIZE*YSIZE);
double *weights_h_d = NULL;
hipMalloc(&weights_h_d, XSIZE*YSIZE);
double *h_out_d = NULL;
hipMalloc(&h_out_d, XSIZE*YSIZE);
double *outputs_d = NULL;
hipMalloc(&outputs_d, XSIZE*YSIZE);
int inputs = 1;
int samples = 1;
int height = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Test<<<gridBlock,threadBlock>>>(training_in_d,training_out_d,data_range_d,weights_in_d,weights_out_d,weights_h_d,h_out_d,outputs_d,inputs,samples,height);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Test<<<gridBlock,threadBlock>>>(training_in_d,training_out_d,data_range_d,weights_in_d,weights_out_d,weights_h_d,h_out_d,outputs_d,inputs,samples,height);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Test<<<gridBlock,threadBlock>>>(training_in_d,training_out_d,data_range_d,weights_in_d,weights_out_d,weights_h_d,h_out_d,outputs_d,inputs,samples,height);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}