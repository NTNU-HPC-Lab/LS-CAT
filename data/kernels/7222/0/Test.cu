#include "hip/hip_runtime.h"
#include "includes.h"

#define INPUTS 5
#define H_LAYERS 3
#define H_HEIGHT 5
#define OUTPUTS 1
#define BIAS 0
#define alpha_d 0.5

#define DATA_FILE "data_for_training.txt"
#define TEST_FILE "data_for_verify.txt"

#define ITERATIONS 10000

//#define DEBUG

//Weight declarations
double weights_in[INPUTS * H_HEIGHT];// = {.15,.20};
double weights_out[OUTPUTS * H_HEIGHT];// = {.65,.7};
__global__ void Test(double *training_in_d, double *training_out_d, double *data_range_d, double *weights_in_d, double *weights_out_d, double *weights_h_d, double *h_out_d, double *outputs_d, int inputs, int samples, int height){
printf("\nDevice:\n");

printf("Training In: %f\n", training_in_d[40 * inputs + 3]);
printf("Training Out: %f\n", training_out_d[40]);
printf("Data Range: %f\n", data_range_d[5 * 2 + 1]);
printf("Weights In: %f\n", weights_in_d[325]);
printf("Weights Out: %f\n", weights_out_d[50]);
printf("Weights H: %f\n", weights_h_d[5 * height * height + 50]);
printf("H Out: %f\n", h_out_d[5 * height + 50]);
printf("Outputs: %f\n", outputs_d[0]);
}