#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_wout(double * weights_out_d, double *weights_out_delta_d, int bit){

//__shared__ double weights_out_delta_ds[10 * 55];

int tix = threadIdx.x;
int tiy = threadIdx.y;

int offset = OUTPUTS * H_HEIGHT;
//weights_out_delta_ds[tiy*offset+tix] = weights_out_delta_d[tiy*offset+tix];

for(int s=32; s > 0; s>>=1){
//int index = 2 * s * tiy;

if(tiy < s && (tiy+s) < blockDim.y)
weights_out_delta_d[tiy*offset+tix] += weights_out_delta_d[(tiy+s)*offset+tix];

__syncthreads();
}

if(tiy == 0){
weights_out_d[tix] -= (alpha_d * weights_out_delta_d[tix] / (true_sample*55.0));
}
__syncthreads();
weights_out_delta_d[tiy*offset+tix] = 0.0;
}