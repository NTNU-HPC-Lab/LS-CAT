#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "back.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *h_out_d = NULL;
hipMalloc(&h_out_d, XSIZE*YSIZE);
double *weights_out_d = NULL;
hipMalloc(&weights_out_d, XSIZE*YSIZE);
double *weights_h_d = NULL;
hipMalloc(&weights_h_d, XSIZE*YSIZE);
double *weights_in_d = NULL;
hipMalloc(&weights_in_d, XSIZE*YSIZE);
double *outputs_d = NULL;
hipMalloc(&outputs_d, XSIZE*YSIZE);
double *deltas_h_d = NULL;
hipMalloc(&deltas_h_d, XSIZE*YSIZE);
double *deltas_h_new_d = NULL;
hipMalloc(&deltas_h_new_d, XSIZE*YSIZE);
double *deltas_o_d = NULL;
hipMalloc(&deltas_o_d, XSIZE*YSIZE);
double *weights_in_delta_d = NULL;
hipMalloc(&weights_in_delta_d, XSIZE*YSIZE);
double *weights_out_delta_d = NULL;
hipMalloc(&weights_out_delta_d, XSIZE*YSIZE);
double *weights_h_delta_d = NULL;
hipMalloc(&weights_h_delta_d, XSIZE*YSIZE);
int height = YSIZE;
int inputs = 1;
int outputs = 1;
int layers = 1;
double *training_in_d = NULL;
hipMalloc(&training_in_d, XSIZE*YSIZE);
double *training_out_d = NULL;
hipMalloc(&training_out_d, XSIZE*YSIZE);
int sample = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
back<<<gridBlock,threadBlock>>>(h_out_d,weights_out_d,weights_h_d,weights_in_d,outputs_d,deltas_h_d,deltas_h_new_d,deltas_o_d,weights_in_delta_d,weights_out_delta_d,weights_h_delta_d,height,inputs,outputs,layers,training_in_d,training_out_d,sample);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
back<<<gridBlock,threadBlock>>>(h_out_d,weights_out_d,weights_h_d,weights_in_d,outputs_d,deltas_h_d,deltas_h_new_d,deltas_o_d,weights_in_delta_d,weights_out_delta_d,weights_h_delta_d,height,inputs,outputs,layers,training_in_d,training_out_d,sample);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
back<<<gridBlock,threadBlock>>>(h_out_d,weights_out_d,weights_h_d,weights_in_d,outputs_d,deltas_h_d,deltas_h_new_d,deltas_o_d,weights_in_delta_d,weights_out_delta_d,weights_h_delta_d,height,inputs,outputs,layers,training_in_d,training_out_d,sample);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}