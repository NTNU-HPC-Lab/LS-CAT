#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back(double *h_out_d, double *weights_out_d, double *weights_h_d, double *weights_in_d, double *outputs_d, double *deltas_h_d, double *deltas_h_new_d, double *deltas_o_d, double *weights_in_delta_d, double *weights_out_delta_d, double *weights_h_delta_d, int height, int inputs, int outputs, int layers, double *training_in_d, double *training_out_d, int sample){

int i, j;

int tix = threadIdx.x;
int tiy = threadIdx.y + sample;

int h_offset = tiy * layers * height;
int w_o_d_offset = tiy * outputs * height;
int w_h_d_offset = tiy * (layers-1) * height * height;
int w_i_d_offset = tiy * inputs * height;
int d_h_offset = tiy * height;

double delta_sum, temp;

/*__shared__ double h_out_ds[H_LAYERS*H_HEIGHT];
__shared__ double weights_h_ds[(H_LAYERS-1)*H_HEIGHT*H_HEIGHT];
__shared__ double deltas_h_ds[H_HEIGHT];
__shared__ double deltas_h_new_ds[H_HEIGHT];

for(i=0;i<layers;i++)
h_out_ds[tix*height+i] = h_out_d[tix*height+i];
for(i=0;i<layers-1;i++){
for(j=0;j<height;j++)
weights_h_ds[i*height*height + tix*height + j] = weights_h_d[i*height*height + tix*height + j];
}
deltas_h_ds[tix] = deltas_h_d[tix];
deltas_h_new_ds[tix] = deltas_h_new_d[tix];

__syncthreads();
*/
//output layer
if(tix < outputs){
deltas_o_d[tiy * outputs + tix] = (outputs_d[tiy * outputs + tix] - training_out_d[tiy]);
for(i = 0; i < height; i++){
weights_out_delta_d[w_o_d_offset + (tix * height) + i] = deltas_o_d[tiy * outputs + tix] * h_out_d[h_offset + (layers-1)*height+i];
}
}

__syncthreads();

//hidden layer

//layer connected to output
delta_sum = 0;
for(i = 0; i < outputs; i++){
delta_sum += weights_out_d[tix + (i * height)] * deltas_o_d[tiy * outputs + i];
}
temp = h_out_d[h_offset + (layers-1)*height + tix];
deltas_h_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

for(i = 0; i < height; i++){
weights_h_delta_d[w_h_d_offset + (layers-2)*height*height + (tix * height) + i] = deltas_h_d[d_h_offset + tix] * h_out_d[h_offset + (layers-2)*height+i];
}

__syncthreads();

//each hidden layer not connected to input/hidden output layer
for(i = layers - 2; i > 0; i--){
delta_sum = 0;
for(j = 0; j < height; j++){
delta_sum += weights_h_d[i*height*height + j*height + tix] * deltas_h_d[d_h_offset + j];
}
temp = h_out_d[h_offset + i*height + tix];
deltas_h_new_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

for(j = 0; j < height; j++){
weights_h_delta_d[w_h_d_offset + (i-1)*height*height + (tix * height) + j] = (deltas_h_new_d[d_h_offset + tix] * h_out_d[h_offset + (i-1)*height+j]);
}

__syncthreads();
//change pointers to simulate copying memory
deltas_h_d[d_h_offset + tix] = deltas_h_new_d[d_h_offset + tix];

__syncthreads();

}

//Layer connected to inputs
delta_sum = 0;
for(i=0; i<height; i++){
delta_sum += weights_h_d[i*height + tix] * deltas_h_d[d_h_offset + i];
}
temp = h_out_d[h_offset + tix];
deltas_h_new_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

for(i=0; i<inputs; i++){
weights_in_delta_d[w_i_d_offset + tix*inputs+i] = (deltas_h_new_d[d_h_offset + tix] * training_in_d[tiy * inputs + i]);
}

__syncthreads();

}