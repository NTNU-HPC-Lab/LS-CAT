#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void error_reduc(double *error_d, int bit){
__shared__ double error_ds[55];
int tix = threadIdx.x;
error_ds[tix] = error_d[tix];

__syncthreads();

for(int s = 32; s > 0; s>>=1){
//int index = 2 * s * threadIdx.x;

if(tix < s && (tix+s) < true_sample){
error_ds[tix] += error_ds[tix + s];
}

__syncthreads();
}


if(tix == 0){
//printf("GPU Error before divide: %f\n",error_d[0]);
error_ds[tix] /= 55.0;
printf("GPU Error: %f\n", error_ds[tix] * 100.0);
}

error_d[tix] = 0.0;

}