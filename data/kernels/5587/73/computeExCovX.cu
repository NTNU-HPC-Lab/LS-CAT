#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeExCovX(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centr_x, double *centr_y, double *centr_z, double gauss_d1, double gauss_d2, double *e_x_cov_x, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];
double d_x = static_cast<double>(trans_x[pid]);
double d_y = static_cast<double>(trans_y[pid]);
double d_z = static_cast<double>(trans_z[pid]);
double t_x, t_y, t_z;


for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
int vid = voxel_id[j];

t_x = d_x - centr_x[vid];
t_y = d_y - centr_y[vid];
t_z = d_z - centr_z[vid];

e_x_cov_x[j] =  exp(-gauss_d2 * ((t_x * icov00[vid] + t_y * icov01[vid] + t_z * icov02[vid]) * t_x
+ ((t_x * icov10[vid] + t_y * icov11[vid] + t_z * icov12[vid]) * t_y)
+ ((t_x * icov20[vid] + t_y * icov21[vid] + t_z * icov22[vid]) * t_z)) / 2.0);
}
}
}