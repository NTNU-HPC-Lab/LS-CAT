#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointHessian2(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph155, double *ph165, double *ph175)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
__shared__ double h_ang[9];

if (threadIdx.x < 9) {
h_ang[threadIdx.x] = dh_ang[36 + threadIdx.x];
}

__syncthreads();

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];

//Orignal coordinates
double o_x = static_cast<double>(x[pid]);
double o_y = static_cast<double>(y[pid]);
double o_z = static_cast<double>(z[pid]);

ph155[i] = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
ph165[i] = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];
ph175[i] = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];

}
}