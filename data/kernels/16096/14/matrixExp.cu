#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixExp(double *a, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){
c[y * cc + x] = exp(a[y * cc + x]);
}

}