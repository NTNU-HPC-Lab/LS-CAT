#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTanh(double *a, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row

if(x < cc && y < cr){
c[y * cc + x] = tanh(a[y * cc + x]);
}

}