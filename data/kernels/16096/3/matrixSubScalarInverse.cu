#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSubScalarInverse(double *a, double b, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){

c[y * cc + x] = b-a[y * cc + x];
}

}