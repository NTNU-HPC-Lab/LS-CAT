#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTranspose(double *a, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){

for(int i = 0; i<cc; i++) {

c[y * cc + x+i] = a[x * cc + y + i];

}
}


}