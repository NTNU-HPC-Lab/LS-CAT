#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSubScalar(double *a, double b, double *c, int cr, int cc){

int x = blockIdx.x * blockDim.x + threadIdx.x; // col
int y = blockIdx.y * blockDim.y + threadIdx.y; // row


if(x < cc && y < cr){

c[y * cc + x] = a[y * cc + x]-b;
}

}