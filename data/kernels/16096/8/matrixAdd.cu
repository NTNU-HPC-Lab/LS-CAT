#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd(double *a, double *b, double *c, int cr, int cc){

long x = blockIdx.x * blockDim.x + threadIdx.x; // col
long y = blockIdx.y * blockDim.y + threadIdx.y; // row

if(x < cc && y < cr){
c[y * cc + x] = a[y * cc + x] + b[y * cc + x];
}

}