#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;

printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
" %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
ix, iy, idx, A[idx]);
}