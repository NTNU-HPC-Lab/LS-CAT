#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

if (ix < nx && iy < ny)
{
out[ix * ny + iy] = in[iy * nx + ix];
}
}