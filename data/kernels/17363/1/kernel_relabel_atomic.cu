#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_relabel_atomic(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1)
{
int x1 = threadIdx.x;
int y1 = threadIdx.y;
int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
int thid = __umul24(y, width1) + x;

__shared__ int height_fn[356];


int temp_mult = __umul24(y1 + 1, 34) + x1 + 1, temp_mult1 = __umul24(y1, 32) + x1;

height_fn[temp_mult] = g_graph_height[thid];

(threadIdx.x == 31 && x < width1 - 1) ? height_fn[temp_mult + 1] = (g_graph_height[thid + 1]) : 0;
(threadIdx.x == 0 && x > 0) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
(threadIdx.y == 7 && y < rows1 - 1) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
(threadIdx.y == 0 && y > 0) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

__syncthreads();


int min_flow_pushed = g_left_weight[thid];
int flow_push = g_push_reser[thid];

if (flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
g_relabel_mask[thid] = 2;
else
{
(flow_push > 0 && (((height_fn[temp_mult] == height_fn[temp_mult - 1] + 1) && g_left_weight[thid] > 0) || ((height_fn[temp_mult] == height_fn[temp_mult + 1] + 1) && g_right_weight[thid] > 0) || ((height_fn[temp_mult] == height_fn[temp_mult + 34] + 1) && g_down_weight[thid] > 0) || ((height_fn[temp_mult] == height_fn[temp_mult - 34] + 1) && g_up_weight[thid] > 0) || (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0))) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0;
}


__syncthreads();

if (thid < graph_size1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
{
if (g_sink_weight[thid] > 0)
{
g_height_write[thid] = 1;
}
else
{
int min_height = graph_size;
(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1]) ? min_height = height_fn[temp_mult - 1] : 0;
(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0;
(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34]) ? min_height = height_fn[temp_mult + 34] : 0;
(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34]) ? min_height = height_fn[temp_mult - 34] : 0;
g_height_write[thid] = min_height + 1;
}
}

}