#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_push_atomic2( int *g_terminate, int *g_push_reser, int *s_push_reser, int *g_block_num, int width1)
{

int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
int thid = __umul24( y , width1 ) + x ;

if( s_push_reser[thid] - g_push_reser[thid] != 0)
{
g_terminate[blockIdx.y * (*g_block_num) + blockIdx.x] = 1 ;
}

}