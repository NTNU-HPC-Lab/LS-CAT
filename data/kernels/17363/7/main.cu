#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kernel_push2_stochastic.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *g_left_weight = NULL;
hipMalloc(&g_left_weight, XSIZE*YSIZE);
int *g_right_weight = NULL;
hipMalloc(&g_right_weight, XSIZE*YSIZE);
int *g_down_weight = NULL;
hipMalloc(&g_down_weight, XSIZE*YSIZE);
int *g_up_weight = NULL;
hipMalloc(&g_up_weight, XSIZE*YSIZE);
int *g_sink_weight = NULL;
hipMalloc(&g_sink_weight, XSIZE*YSIZE);
int *g_push_reser = NULL;
hipMalloc(&g_push_reser, XSIZE*YSIZE);
int *g_pull_left = NULL;
hipMalloc(&g_pull_left, XSIZE*YSIZE);
int *g_pull_right = NULL;
hipMalloc(&g_pull_right, XSIZE*YSIZE);
int *g_pull_down = NULL;
hipMalloc(&g_pull_down, XSIZE*YSIZE);
int *g_pull_up = NULL;
hipMalloc(&g_pull_up, XSIZE*YSIZE);
int *g_relabel_mask = NULL;
hipMalloc(&g_relabel_mask, XSIZE*YSIZE);
int *g_graph_height = NULL;
hipMalloc(&g_graph_height, XSIZE*YSIZE);
int *g_height_write = NULL;
hipMalloc(&g_height_write, XSIZE*YSIZE);
int graph_size = XSIZE*YSIZE;
int width = XSIZE;
int rows = XSIZE;
int graph_size1 = XSIZE*YSIZE;
int width1 = XSIZE;
int rows1 = 1;
int *d_relabel = NULL;
hipMalloc(&d_relabel, XSIZE*YSIZE);
int *d_stochastic = NULL;
hipMalloc(&d_stochastic, XSIZE*YSIZE);
int *d_counter = NULL;
hipMalloc(&d_counter, XSIZE*YSIZE);
bool *d_finish = NULL;
hipMalloc(&d_finish, XSIZE*YSIZE);
int *g_block_num = NULL;
hipMalloc(&g_block_num, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_push2_stochastic<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_sink_weight,g_push_reser,g_pull_left,g_pull_right,g_pull_down,g_pull_up,g_relabel_mask,g_graph_height,g_height_write,graph_size,width,rows,graph_size1,width1,rows1,d_relabel,d_stochastic,d_counter,d_finish,g_block_num);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kernel_push2_stochastic<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_sink_weight,g_push_reser,g_pull_left,g_pull_right,g_pull_down,g_pull_up,g_relabel_mask,g_graph_height,g_height_write,graph_size,width,rows,graph_size1,width1,rows1,d_relabel,d_stochastic,d_counter,d_finish,g_block_num);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kernel_push2_stochastic<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_sink_weight,g_push_reser,g_pull_left,g_pull_right,g_pull_down,g_pull_up,g_relabel_mask,g_graph_height,g_height_write,graph_size,width,rows,graph_size1,width1,rows1,d_relabel,d_stochastic,d_counter,d_finish,g_block_num);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}