#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_push1_start_stochastic(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish)
{
int x1 = threadIdx.x;
int y1 = threadIdx.y;
int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
int thid = __umul24(y, width1) + x;

__shared__ int height_fn[356];


int temp_mult = __umul24(y1 + 1, 34) + x1 + 1, temp_mult1 = __umul24(y1, 32) + x1;

height_fn[temp_mult] = g_graph_height[thid];

(threadIdx.x == 31 && x < width1 - 1) ? height_fn[temp_mult + 1] = (g_graph_height[thid + 1]) : 0;
(threadIdx.x == 0 && x > 0) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
(threadIdx.y == 7 && y < rows1 - 1) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
(threadIdx.y == 0 && y > 0) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

__syncthreads();

int flow_push = 0, min_flow_pushed = 0;
flow_push = g_push_reser[thid];


if (thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
{
int temp_weight = 0;


temp_weight = g_sink_weight[thid];
min_flow_pushed = flow_push;

if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
{
(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
temp_weight = temp_weight - min_flow_pushed;
g_sink_weight[thid] = temp_weight;
atomicSub(&g_push_reser[thid], min_flow_pushed);

flow_push = flow_push - min_flow_pushed;
}



}


__syncthreads();
min_flow_pushed = g_left_weight[thid];

(flow_push > 0 && (((height_fn[temp_mult] == height_fn[temp_mult - 1] + 1) && min_flow_pushed > 0) || ((height_fn[temp_mult] == height_fn[temp_mult + 1] + 1) && g_right_weight[thid] > 0) || ((height_fn[temp_mult] == height_fn[temp_mult + 34] + 1) && g_down_weight[thid] > 0) || ((height_fn[temp_mult] == height_fn[temp_mult - 34] + 1) && g_up_weight[thid] > 0) || (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0))) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0;



if (thid < graph_size1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
{
if (g_sink_weight[thid] > 0)
{
g_height_write[thid] = 1;
}
else
{
int min_height = graph_size;
(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1]) ? min_height = height_fn[temp_mult - 1] : 0;
(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0;
(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34]) ? min_height = height_fn[temp_mult + 34] : 0;
(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34]) ? min_height = height_fn[temp_mult - 34] : 0;
g_height_write[thid] = min_height + 1;
}
}

}