#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_push1_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1 )
{
int x1 = threadIdx.x ;
int y1 = threadIdx.y ;
int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
int thid = __umul24( y , width1 ) + x ;

__shared__ int height_fn[356];


int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

height_fn[temp_mult] = g_graph_height[thid] ;

(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

__syncthreads();

int flow_push = 0, min_flow_pushed = 0 ;
flow_push = g_push_reser[thid] ;


if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
{
int temp_weight = 0;


temp_weight = g_sink_weight[thid] ;
min_flow_pushed = flow_push ;

if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
{
(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
temp_weight = temp_weight - min_flow_pushed ;
g_sink_weight[thid] = temp_weight ;
atomicSub(&g_push_reser[thid] , min_flow_pushed);
}


flow_push = g_push_reser[thid] ;

min_flow_pushed = flow_push ;

temp_weight = g_left_weight[thid] ;

if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
{
(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
temp_weight = temp_weight - min_flow_pushed ;
atomicSub(&g_left_weight[thid] , min_flow_pushed);
atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
atomicSub(&g_push_reser[thid] , min_flow_pushed);
atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

}else atomicSub(&g_pull_left[thid-1], 1) ;

flow_push = g_push_reser[thid] ;
min_flow_pushed = flow_push ;
temp_weight = g_up_weight[thid] ;

if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
{
(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
temp_weight = temp_weight - min_flow_pushed ;

atomicSub(&g_up_weight[thid] , min_flow_pushed);
atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
atomicSub(&g_push_reser[thid] , min_flow_pushed);
atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

} else atomicSub(&g_pull_up[thid - width1] , 1) ;


flow_push = g_push_reser[thid] ;
min_flow_pushed = flow_push ;
temp_weight = g_right_weight[thid] ;

if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
{
(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
temp_weight = temp_weight - min_flow_pushed ;
atomicSub(&g_right_weight[thid] , min_flow_pushed);
atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
atomicSub(&g_push_reser[thid] , min_flow_pushed);
atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
}else atomicSub( &g_pull_right[thid + 1], 1) ;


flow_push = g_push_reser[thid] ;

min_flow_pushed = flow_push ;
temp_weight = g_down_weight[thid] ;


if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
{
(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
temp_weight = temp_weight - min_flow_pushed ;
atomicSub(&g_down_weight[thid] , min_flow_pushed);
atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
atomicSub(&g_push_reser[thid] , min_flow_pushed);
atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
}else atomicSub( &g_pull_down[thid+width1], 1) ;
}

}