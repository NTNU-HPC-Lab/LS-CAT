#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_bfs(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_graph_height, bool *g_pixel_mask, int vertex_num, int width, int height, int vertex_num1, int width1, int height1, bool *g_over, int *g_counter)
{
/*******************************
*threadId is calculated ******
*****************************/

int thid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

if (thid < vertex_num && g_pixel_mask[thid] == true)
{
int col = thid % width1, row = thid / width1;

if (col < width - 1 && col > 0 && row < height - 1 && row > 0)
{
int height_l = 0, height_d = 0, height_u = 0, height_r = 0;
height_r = g_graph_height[thid + 1];
height_l = g_graph_height[thid - 1];
height_d = g_graph_height[thid + width1];
height_u = g_graph_height[thid - width1];

if (((height_l == (*g_counter) && g_right_weight[thid - 1] > 0)) || ((height_d == (*g_counter) && g_up_weight[thid + width1] > 0) || (height_r == (*g_counter) && g_left_weight[thid + 1] > 0) || (height_u == (*g_counter) && g_down_weight[thid - width1] > 0)))
{
g_graph_height[thid] = (*g_counter) + 1;
g_pixel_mask[thid] = false;
*g_over = true;
}
}
}
}