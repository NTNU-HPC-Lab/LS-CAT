#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kernel_bfs.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *g_left_weight = NULL;
hipMalloc(&g_left_weight, XSIZE*YSIZE);
int *g_right_weight = NULL;
hipMalloc(&g_right_weight, XSIZE*YSIZE);
int *g_down_weight = NULL;
hipMalloc(&g_down_weight, XSIZE*YSIZE);
int *g_up_weight = NULL;
hipMalloc(&g_up_weight, XSIZE*YSIZE);
int *g_graph_height = NULL;
hipMalloc(&g_graph_height, XSIZE*YSIZE);
bool *g_pixel_mask = NULL;
hipMalloc(&g_pixel_mask, XSIZE*YSIZE);
int vertex_num = 1;
int width = XSIZE;
int height = YSIZE;
int vertex_num1 = 1;
int width1 = XSIZE;
int height1 = YSIZE;
bool *g_over = NULL;
hipMalloc(&g_over, XSIZE*YSIZE);
int *g_counter = NULL;
hipMalloc(&g_counter, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_bfs<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_graph_height,g_pixel_mask,vertex_num,width,height,vertex_num1,width1,height1,g_over,g_counter);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kernel_bfs<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_graph_height,g_pixel_mask,vertex_num,width,height,vertex_num1,width1,height1,g_over,g_counter);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kernel_bfs<<<gridBlock,threadBlock>>>(g_left_weight,g_right_weight,g_down_weight,g_up_weight,g_graph_height,g_pixel_mask,vertex_num,width,height,vertex_num1,width1,height1,g_over,g_counter);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}