#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_bfs_t(int *g_push_reser, int  *g_sink_weight, int *g_graph_height, bool *g_pixel_mask, int vertex_num, int width, int height, int vertex_num1, int width1, int height1)
{

int thid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

if (thid < vertex_num && g_pixel_mask[thid] == true)
{
int col = thid % width1, row = thid / width1;

if (col > 0 && row > 0 && col < width - 1 && row < height - 1 && g_push_reser[thid] > 0)
{
g_graph_height[thid] = 1;
g_pixel_mask[thid] = false;
}
else
if (g_sink_weight[thid] > 0)
{
g_graph_height[thid] = -1;
g_pixel_mask[thid] = false;
}
}
}