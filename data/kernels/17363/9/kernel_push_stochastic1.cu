#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_push_stochastic1(int *g_push_reser, int *s_push_reser, int *g_count_blocks, bool *g_finish, int *g_block_num, int width1)
{
int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
int thid = __umul24(y, width1) + x;

s_push_reser[thid] = g_push_reser[thid];

if (thid == 0)
{
if ((*g_count_blocks) == 0)
(*g_finish) = false;
}
}