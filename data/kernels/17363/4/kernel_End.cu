#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_End(int *g_stochastic, int *g_count_blocks, int *g_counter)
{
int thid = blockIdx.x * blockDim.x + threadIdx.x;
if (thid < (*g_counter))
{
if (g_stochastic[thid] == 1)
atomicAdd(g_count_blocks, 1);
//(*g_count_blocks) = (*g_count_blocks) + 1 ;
}
}