#include "hip/hip_runtime.h"
#include "includes.h"


#define INTERVALS 1000000

// Max number of threads per block
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

// Synchronous error checking call. Enable with nvcc -DDEBUG
__global__ void integrateSimple(float *sum, float step, int threads, int blocks)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;

for (int i = idx; i < INTERVALS; i+=threads*blocks)
{
float x = (i+0.5f) * step;
sum[idx] += 4.0f / (1.0f+ x*x);
}
}