#include "hip/hip_runtime.h"
#include "includes.h"


#define INTERVALS 1000000

// Max number of threads per block
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

// Synchronous error checking call. Enable with nvcc -DDEBUG
__global__ void integrateOptimised(int *n, float *g_sum)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
int tx = threadIdx.x;

// Shared memory to hold the sum for each block
__shared__ float s_sum[THREADS];

float sum = 0.0f;
float step  = 1.0f / (float)*n;

for (int i = idx + 1; i <= *n; i += blockDim.x * BLOCKS)
{
float x = step * ((float)i - 0.5f);
sum += 4.0f / (1.0f+ x*x);
}
s_sum[tx] = sum * step;

// Wait for all threads to catch up
__syncthreads();

// For each block, do sum using shared memory
for (int i = blockDim.x / 2; i > 0; i >>= 1)
{
if (tx < i)
{
s_sum[tx] += s_sum[tx + i];
}

__syncthreads();
}

// Write results to global memory
g_sum[idx] = s_sum[tx];
}