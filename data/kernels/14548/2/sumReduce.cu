#include "hip/hip_runtime.h"
#include "includes.h"


#define INTERVALS 1000000

// Max number of threads per block
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

// Synchronous error checking call. Enable with nvcc -DDEBUG
__global__ static void sumReduce(int *n, float *g_sum)
{
int tx = threadIdx.x;
__shared__ float s_sum[THREADS];

if (tx < BLOCKS)
s_sum[tx] = g_sum[tx * THREADS];
else
{
s_sum[tx] = 0.0f;
}

// For each block
for (int i = blockDim.x / 2; i > 0; i >>= 1)
{
if (tx < i)
{
s_sum[tx] += s_sum[tx + i];
}
__syncthreads();
}

g_sum[tx] = s_sum[tx];
}