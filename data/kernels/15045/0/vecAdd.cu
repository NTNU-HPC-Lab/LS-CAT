#include "hip/hip_runtime.h"
#include "includes.h"
/**********************************************************
* @author  Pulkit Verma
* @email   technopreneur[dot]pulkit[at]gmail[dot]com
**********************************************************/

// The program takes two equal size vectors as input and outputs their vector sum


__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if(i<len)
out[i]=in1[i]+in2[i];

return;
}