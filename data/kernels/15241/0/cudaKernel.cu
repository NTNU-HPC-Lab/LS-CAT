#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernel(int n, double* gpuWeights, int* gpuG, int* gpuTempGrid, int *flag)
{
// Moment's coordinates in the grid //
// allocate shared memory for weights
int momentCol = blockIdx.x*blockDim.x + threadIdx.x;
int	momentRow = blockIdx.y*blockDim.y + threadIdx.y;

int gridRowIdx, gridColIdx;
// Variable storing the total neighbourhood influence //
double weightFactor = 0.0;
// Each thread calculates the spin for a block of moments //
// The step is based on the GRID_SIZE and BLOCK_SIZE //
for(int i=momentRow; i<n; i+=blockDim.y*gridDim.y)
{
for(int j=momentCol; j<n; j+=blockDim.x*gridDim.x)
{
weightFactor = 0.0;
// Read 24 neighbours of every moment and calculate their total influence //
for(int weightsRow=0; weightsRow<5; weightsRow++)
{
for(int weightsCol=0; weightsCol<5; weightsCol++)
{
if(weightsCol==2 && weightsRow==2)
continue;
// Calculate neighbour's coordinates in G //
// using modulus to satisfy boundary conditions //
gridRowIdx = (weightsRow - 2 + i + n) % n;
gridColIdx = (weightsCol - 2 + j + n) % n;

weightFactor+= gpuG[gridRowIdx * n + gridColIdx] * gpuWeights[weightsRow*5+weightsCol];
}
}
// Update moment's atomic spin //
// Set flag if a spin value transition has been done //
if(weightFactor < 0.0001 && weightFactor > -0.0001)
{
gpuTempGrid[n*i+j] = gpuG[n*i+j];
}else if(weightFactor > 0.00001)
{
gpuTempGrid[n*i+j] = 1;
if (gpuG[n*i+j] == -1)
{
*flag = 1;
}
}else
{
gpuTempGrid[n*i+j] = -1;
if (gpuG[n*i+j] == -1)
{
*flag = 1;
}
}
}
}
}