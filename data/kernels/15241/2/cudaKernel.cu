#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernel(int n, double* gpuWeights, int* gpuG, int* gpuTempGrid, int* flag)
{
// Moment's coordinates in the grid //
int momentRow = blockIdx.y*blockDim.y + threadIdx.y;
int momentCol = blockIdx.x*blockDim.x + threadIdx.x;
int gridRowIdx, gridColIdx;

// Variable storing the total neighbourhood influence //
double weightFactor = 0.0;

// Check if coordinates are valid //
if(momentRow < n && momentCol < n){
// Read 24 neighbours of every moment and calculate their total influence //
for(int row=0; row<5; row++)
{
for(int col=0; col<5; col++)
{
if(row==2 && col==2)
continue;
// Calculate neighbour's coordinates in G //
// using modulus to satisfy boundary conditions //
gridRowIdx = (row - 2 + momentRow + n) % n;
gridColIdx = (col - 2 + momentCol + n) % n;

weightFactor+= gpuG[gridRowIdx * n + gridColIdx] * gpuWeights[row*5+col];
}
}
// Update moment's atomic spin //
// Set flag if a spin value transition has been done //
if(weightFactor < 0.0001 && weightFactor > -0.0001)
{
gpuTempGrid[n*momentRow+momentCol] = gpuG[n*momentRow+momentCol];
}else if(weightFactor > 0.00001)
{
gpuTempGrid[n*momentRow+momentCol] = 1;
if (gpuG[n*momentRow+momentCol] == -1)
{
*flag = 1;
}
}else
{
gpuTempGrid[n*momentRow+momentCol] = -1;
if (gpuG[n*momentRow+momentCol] == 1)
{
*flag = 1;
}
}
}
}