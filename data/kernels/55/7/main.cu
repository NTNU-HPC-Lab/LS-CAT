#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tonemap.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_x = NULL;
hipMalloc(&d_x, XSIZE*YSIZE);
float *d_y = NULL;
hipMalloc(&d_y, XSIZE*YSIZE);
float *d_log_Y = NULL;
hipMalloc(&d_log_Y, XSIZE*YSIZE);
float *d_cdf_norm = NULL;
hipMalloc(&d_cdf_norm, XSIZE*YSIZE);
float *d_r_new = NULL;
hipMalloc(&d_r_new, XSIZE*YSIZE);
float *d_g_new = NULL;
hipMalloc(&d_g_new, XSIZE*YSIZE);
float *d_b_new = NULL;
hipMalloc(&d_b_new, XSIZE*YSIZE);
float min_log_Y = 1;
float max_log_Y = 1;
float log_Y_range = 1;
int num_bins = 1;
int num_pixels_y = 1;
int num_pixels_x = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
tonemap<<<gridBlock,threadBlock>>>(d_x,d_y,d_log_Y,d_cdf_norm,d_r_new,d_g_new,d_b_new,min_log_Y,max_log_Y,log_Y_range,num_bins,num_pixels_y,num_pixels_x);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
tonemap<<<gridBlock,threadBlock>>>(d_x,d_y,d_log_Y,d_cdf_norm,d_r_new,d_g_new,d_b_new,min_log_Y,max_log_Y,log_Y_range,num_bins,num_pixels_y,num_pixels_x);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
tonemap<<<gridBlock,threadBlock>>>(d_x,d_y,d_log_Y,d_cdf_norm,d_r_new,d_g_new,d_b_new,min_log_Y,max_log_Y,log_Y_range,num_bins,num_pixels_y,num_pixels_x);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}