#include "hip/hip_runtime.h"
#include "includes.h"
#define _CRT_SECURE_NO_WARNINGS

#define BLOCK_DIM 16


__global__ void transposition(int* matrix, int* matrixOut, int length, int width)
{
__shared__ int tempMatrix[BLOCK_DIM][BLOCK_DIM];//ðàçäåëÿåìàÿ ïàìÿòü

int temp;

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

if ((i < length) && (j < width))
{
temp = j * length + i;
tempMatrix[threadIdx.y][threadIdx.x] = matrix[temp];
}

__syncthreads();

i = blockIdx.y * blockDim.y + threadIdx.x;//èíäåêñ áëîêà, ðàçìåðíîñòü áëîêà, èíäåêñ ïîòîêà
j = blockIdx.x * blockDim.x + threadIdx.y;

if ((i < width) && (j < length))
{
temp = j * width + i;
matrixOut[temp] = tempMatrix[threadIdx.x][threadIdx.y];
}
}