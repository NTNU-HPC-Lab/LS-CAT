#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_double(double* A, double* B ,  double* C, int M, int N, int K)
{



int bx = blockIdx.x ;
int by = blockIdx.y ;

int tx = threadIdx.x ;
int ty = threadIdx.y ;

int row = by * TILE_WIDTH + ty ;
int col = bx * TILE_WIDTH + tx ;

__shared__ double SA[TILE_WIDTH][TILE_WIDTH+1] ;
__shared__ double SB[TILE_WIDTH][TILE_WIDTH+1] ;

double Csub = 0;



for (int i = 0; i < (K-1)/TILE_WIDTH +1 ; ++i)
{
/* code */


if ( (row < M) && (i * TILE_WIDTH + tx < K ) ){
SA[ty][tx] = A[row*K + i * TILE_WIDTH + tx] ;
}
else{
SA[ty][tx] = 0;
}




if ( (col < N ) &&  ( i * TILE_WIDTH + ty < K)){
SB[tx][ty] = B[ col * K + i*TILE_WIDTH + ty] ;
}
else{
SB[tx][ty] = 0;
}


__syncthreads() ;

for (int k = 0; k < TILE_WIDTH; ++k){
Csub += SA[ty][k]*SB[tx][k] ;
}

__syncthreads() ;


}

//C[row*n + col] = Csub ;

if ( (row < M ) && ( col < N )){
C[ row * N + col] = Csub ;
}



}