#include "hip/hip_runtime.h"
#include "includes.h"
/* objective
* 	C = A*B  // A[m][k], B[k][n], C[m][n]
* compile: nvcc --gpu-architecture=compute_60 --gpu-code=sm_60 -O3 matmul_double.cu -o matmul_double
Using nvprof for this lab

nvprof -- query-metrics
nvprof dram_read_transactions ./test 1024 1024  128
nvprof ./test 1024 1024 128

second line of result shows time for GPU kernel

GFlop   ( 2MNK * 10^-9 ) / time (second)

*/



#define TILE_WIDTH 16

__global__ void matmul_double(double* A, double* B , double* C, int M, int N, int K)
{
/// complete code

int bx = blockIdx.x ;
int by = blockIdx.y ;

int tx = threadIdx.x ;
int ty = threadIdx.y ;

int row = by * TILE_WIDTH + ty ;
int col = bx * TILE_WIDTH + tx ;

__shared__ double SA[TILE_WIDTH][TILE_WIDTH] ;
__shared__ double SB[TILE_WIDTH][TILE_WIDTH] ;

double Csub = 0;

for (int i = 0; i < (K-1)/TILE_WIDTH +1 ; ++i)
{
/* code */
//SA[ty][tx] = A[row*n + i * TILE_WIDTH + tx] ;
//SB[ty][tx] = B[(i * TILE_WIDTH + ty )*n + col   ] ;

if ( (row < M) && (i * TILE_WIDTH + tx < K ) ){
SA[ty][tx] = A[row*K + i * TILE_WIDTH + tx] ;
}
else{
SA[ty][tx] = 0;
}

if ( (col < N ) && ( i * TILE_WIDTH + ty < K) ){
SB[ty][tx] = B[(i*TILE_WIDTH + ty)*N + col] ;
}
else{
SB[ty][tx] = 0;
}



__syncthreads() ;

for (int k = 0; k < TILE_WIDTH; ++k){
Csub += SA[ty][k] * SB[k][tx] ;
}

__syncthreads() ;


}

//C[row*n + col] = Csub ;

if ( (row < M ) && ( col < N )){
C[ row * N + col] = Csub ;
}



}