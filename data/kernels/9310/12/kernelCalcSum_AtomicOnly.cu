#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelCalcSum_AtomicOnly(const int* dataArray, int arraySize, int* sum)
{
int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
atomicAdd(sum, dataArray[arrayIndex]);
}
}