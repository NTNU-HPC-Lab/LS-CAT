#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax1(const int* dataArray, int arraySize, int* maxVal)
{
int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
atomicMax(maxVal, dataArray[arrayIndex]);
}
}