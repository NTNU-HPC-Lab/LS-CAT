#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax5(const int* dataArray, int arraySize, int* maxVal)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex1 = (int)(blockDim.x * blockIdx.x + threadIdx.x);
int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;

cache[cacheIndex] = INT_MIN;

if (arrayIndex1 < arraySize)
{
cache[cacheIndex] = max(cache[cacheIndex] , dataArray[arrayIndex1]);
}

if (arrayIndex2 < arraySize)
{
cache[cacheIndex] = max(cache[cacheIndex] , dataArray[arrayIndex2]);
}

__syncthreads();

int blockSize = blockDim.x;
for (int offset = blockSize >> 1; offset > 32; offset >>= 1)
{
if (cacheIndex < offset)
{
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ offset]);
}
__syncthreads();
}

// ワープは32スレッド単位なので、スレッドIDが32未満になったところでループ内容を展開
if (threadIdx.x < 32)
{
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 32]);
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 16]);
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 8]);
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 4]);
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 2]);
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ 1]);
}

if (cacheIndex == 0)
{
atomicMax(maxVal, cache[0]);
}
}