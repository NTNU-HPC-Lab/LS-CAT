#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax3(const int* dataArray, int arraySize, int* maxVal)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
cache[cacheIndex] = dataArray[arrayIndex];
}
else
{
cache[cacheIndex] = INT_MIN;
}

__syncthreads();

int blockSize = blockDim.x;
for (int offset = blockSize >> 1; offset > 0; offset >>= 1) // for文の中身を変更
{
if (cacheIndex < offset)
{
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ offset]); // オフセット計算も+からxorに変更（offsetは2の累乗値なのでxorにしても加算と同じになる）
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicMax(maxVal, cache[0]);
}
}