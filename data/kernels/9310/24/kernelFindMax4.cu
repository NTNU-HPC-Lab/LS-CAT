#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax4(const int* dataArray, int arraySize, int* maxVal)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex1 = (int)(blockDim.x * blockIdx.x + threadIdx.x); // グローバルメモリの1つ目の要素番号
int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;         // グローバルメモリの2つ目の要素番号

cache[cacheIndex] = INT_MIN;

if (arrayIndex1 < arraySize)
{
cache[cacheIndex] = max(cache[cacheIndex] , dataArray[arrayIndex1]);    // シェアードメモリと比較
}

if (arrayIndex2 < arraySize)
{
cache[cacheIndex] = max(cache[cacheIndex] , dataArray[arrayIndex2]);    // シェアードメモリと比較
}

__syncthreads();

int blockSize = blockDim.x;
for (int offset = blockSize >> 1; offset > 0; offset >>= 1)
{
if (cacheIndex < offset)
{
cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ offset]);
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicMax(maxVal, cache[0]);
}
}