#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelCalcSum_EffectiveShareAccess(const int* dataArray, int arraySize, int* sum)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
cache[cacheIndex] = dataArray[arrayIndex];
}
else
{
cache[cacheIndex] = 0;
}

__syncthreads();

int blockSize = blockDim.x;
for (int offset = blockSize >> 1; offset > 0; offset >>= 1) // code in this for block is changed
{
if (cacheIndex < offset)
{
cache[cacheIndex] += cache[cacheIndex ^ offset];
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicAdd(sum, cache[0]);
}
}