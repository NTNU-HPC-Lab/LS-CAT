#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax2(const int* dataArray, int arraySize, int* maxVal)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
cache[cacheIndex] = dataArray[arrayIndex];
}
else
{
cache[cacheIndex] = INT_MIN;
}

__syncthreads();

int baseIndex = threadIdx.x * 2;
int blockSize = blockDim.x;
for (int offset = 1; offset < blockSize; offset *= 2)
{
if (baseIndex + offset < blockSize)
{
cache[baseIndex] = max(cache[baseIndex], cache[baseIndex + offset]);
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicMax(maxVal, cache[0]);
}
}