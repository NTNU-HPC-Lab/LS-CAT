#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelCalcSum_ShareMem(const int* dataArray, int arraySize, int* sum)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex = (int)(blockDim.x * blockIdx.x + threadIdx.x);
if (arrayIndex < arraySize)
{
cache[cacheIndex] = dataArray[arrayIndex];
}
else
{
cache[cacheIndex] = 0;
}

__syncthreads();

int baseIndex = threadIdx.x * 2;
int blockSize = blockDim.x;
for (int offset = 1; offset < blockSize; offset *= 2)
{
if (baseIndex + offset < blockSize)
{
cache[baseIndex] += cache[baseIndex + offset];
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicAdd(sum, cache[0]);
}
}