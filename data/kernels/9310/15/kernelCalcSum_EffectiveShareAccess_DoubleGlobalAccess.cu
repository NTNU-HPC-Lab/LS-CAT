#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelCalcSum_EffectiveShareAccess_DoubleGlobalAccess(const int* dataArray, int arraySize, int* sum)
{
__shared__ extern int cache[];

int cacheIndex = threadIdx.x;

int arrayIndex1 = (int)(blockDim.x * blockIdx.x + threadIdx.x); // first element
int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;         // second element

cache[cacheIndex] = 0;

if (arrayIndex1 < arraySize)
{
cache[cacheIndex] += dataArray[arrayIndex1];
}

if (arrayIndex2 < arraySize)
{
cache[cacheIndex] += dataArray[arrayIndex2];
}

__syncthreads();

int blockSize = blockDim.x;
for (int offset = blockSize >> 1; offset > 0; offset >>= 1)
{
if (cacheIndex < offset)
{
cache[cacheIndex] += cache[cacheIndex ^ offset];
}
__syncthreads();
}

if (cacheIndex == 0)
{
atomicAdd(sum, cache[0]);
}
}