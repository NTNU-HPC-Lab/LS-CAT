#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
// use threadIdx.x to access thread index
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}