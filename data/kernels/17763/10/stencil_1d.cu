#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out) {
// within a block, threads share data via shared memory ("global memory")
// data is not visible to threads in other blocks
// use __shared__ to declare a var/array in shared memory

__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
// each thread processs one output element (blockDim.x elements per block)
int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
int lindex = threadIdx.x + RADIUS;

// read input elements into shared memory
temp[lindex] = in[gindex];
if (threadIdx.x < RADIUS) {
temp[lindex - RADIUS] = in[gindex - RADIUS];
temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
}

// synchronize all threads in the block : ensure all data is available
__syncthreads();

// apply the stencil
int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++) {
result += temp[lindex + offset];
}

// store the result
out[gindex-RADIUS] = result;
}