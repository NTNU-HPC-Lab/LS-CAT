#include "hip/hip_runtime.h"
#include "includes.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if((i>=max_x) || (j>=max_y)) return;
int pixel_index = j*max_x + i;
// each threads gets same seed, a different sequence number and no offset
hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}