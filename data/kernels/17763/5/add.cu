#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
// each block handles a different element of the array
// on the device, each block can execute in parallel
// use blockIdx.x to access block index
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}