#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int n)
{
//blockDim.x represents threads per block
int index = threadIdx.x + blockIdx.x * blockDim.x;
// as we need to avoid to go beyond the end of the arrays, we need to define the limit
if (index < n)
c[index] = a[index] + b[index];
}