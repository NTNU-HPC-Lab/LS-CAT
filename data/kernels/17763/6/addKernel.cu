#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int* c, const int* a, const int* b, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
// since we're asking for one more thread than elements in the arrays
// we need to handle size to make sure we don't access beyond the end of the array
if (i < size) {
c[i] = a[i] + b[i];
}
}