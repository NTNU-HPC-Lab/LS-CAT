#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
// Assumes values in 'p' are either 1 or 0
// Assumes s[0:31] are allocated
// Sums p across warp, returning the result. Suggest you put
// result in s[0] and return it
// You may change any value in s
// You should execute no more than 5 + operations (if you're doing
// 31, you're doing it wrong)
//
// TODO: Fill in the rest of this function

return s[0];
}
__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
extern __shared__ unsigned int s[];
int t = threadIdx.x;
int p = d_in[t];
unsigned int sr = shared_reduce(p, s);
if (t == 0)
{
*d_out_shared = sr;
}
}