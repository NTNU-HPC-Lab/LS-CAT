#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
int myId = threadIdx.x + blockDim.x * blockIdx.x;
int myItem = d_in[myId];
int myBin = myItem % BIN_COUNT;
atomicAdd(&(d_bins[myBin]), 1);
}