#include "hip/hip_runtime.h"
#include "includes.h"

#define B 1 // blocks in the grid
#define T 10 // threads in a block


#ifdef BAMBOO_PROFILING
#else
#endif


__global__ void gpu_mult(int *a,int *b, int *c, int N) {

int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if( col < N && row < N) {
for(int i = 0; i < N; i++) {
sum += a[row * N + i] * b[i * N + col];
}
c[row * N + col] = sum;
}
}