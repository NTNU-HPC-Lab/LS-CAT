#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div180(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx ; i += 2){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){

posisi = matrix[Index + 1] * nx + matrix[Index];
atomicAdd(&newMatrix[posisi],1);

posisi = matrix[Index + (nx + 1)] * nx + matrix[Index + nx];
atomicAdd(&newMatrix[posisi],1);
}
}
}