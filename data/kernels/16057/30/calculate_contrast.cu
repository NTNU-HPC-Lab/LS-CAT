#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_contrast(float *norm,float *contrast,int *dif,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
int tid=threadIdx.x;
//printf("%d\n",tid);
if (idx >= max*max) return;
// in-place reduction in global memory
//float *contrast=norm+blockIdx.x*blockDim.x;
if(idx<size){
contrast[idx]=norm[idx]*dif[idx];
//printf("%f %f\n",norm[idx],contrast[idx]);
__syncthreads();
}
for (int stride = 1; stride < max; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
contrast[idx] += contrast[idx+ stride];
//printf("%d %f\n",idx,contrast[idx]);
}
// synchronize within threadblock
__syncthreads();
}

if (idx == 0){
printf("contrast %f\n",contrast[0]);
}
}