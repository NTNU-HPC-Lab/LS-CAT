#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div0(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx ; i += 2){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){

posisi = matrix[Index] * Max + matrix[Index + 1];
atomicAdd(&newMatrix[posisi],1);

posisi = matrix[Index + Max] * Max + matrix[Index + (Max + 1)];
atomicAdd(&newMatrix[posisi],1);
}
}
}