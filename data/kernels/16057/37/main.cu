#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "calculate_IMC.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *norm = NULL;
hipMalloc(&norm, XSIZE*YSIZE);
float *IMC = NULL;
hipMalloc(&IMC, XSIZE*YSIZE);
float *HX = NULL;
hipMalloc(&HX, XSIZE*YSIZE);
float *HY = NULL;
hipMalloc(&HY, XSIZE*YSIZE);
float *entropy = NULL;
hipMalloc(&entropy, XSIZE*YSIZE);
float *px = NULL;
hipMalloc(&px, XSIZE*YSIZE);
float *py = NULL;
hipMalloc(&py, XSIZE*YSIZE);
float *HXY = NULL;
hipMalloc(&HXY, XSIZE*YSIZE);
int max = 1;
float sum = 1;
int size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_IMC<<<gridBlock,threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
calculate_IMC<<<gridBlock,threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
calculate_IMC<<<gridBlock,threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}