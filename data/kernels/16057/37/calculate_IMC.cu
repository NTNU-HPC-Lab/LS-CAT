#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_IMC(float *norm,float *IMC,float *HX,float *HY,float *entropy,float *px,float *py,float *HXY,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
int tid=threadIdx.x;

int i;
for(i=0;i<max;i++){
if(idx>=i*max && idx<(i+1)*(max) && norm[idx]>0){
HX[idx]=-(norm[idx]*log10f(norm[idx]));
//printf("%d,i %d  %f %f \n",idx,i,miu_x[idx],norm[idx]);
}
}

if(idx<size && norm[idx] !=0){
entropy[idx]=-(norm[idx]*log10f(norm[idx]));
//printf("%d f3 %f \n",idx,entropy[idx]);
__syncthreads();
}



// for(i=0;i<max;i++){
//     if(idx>=i*max && idx<(i+1)*(max) && norm[idx]>0){
//         px[idx]=norm[idx];
//         //printf("%d,i %d  %f %f \n",idx,i,miu_x[idx],norm[idx]);
// }
// }
if(idx<size){
px[idx]=norm[idx];
}

int c=0;
for(i=0;i<max;i++){
// printf("%d",batas);
if(c==i && idx<max){
py[c*max+idx]=norm[idx*max+i];
//printf("%d %d,i %d  %f %f %d \n",idx,idx,i,stdy[idx],norm[idx*max+i],idx*max+i);
c++;
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}


int b=0;
for(i=0;i<max;i++){
// printf("%d",batas);
if(b==i && idx<max &&norm[idx*max+i]>0){
HY[b*max+idx]=-(norm[idx*max+i]*log10f(norm[idx*max+i]));
//printf("%d %d,i %d  %f %f %d \n",idx,idx,i,HY[b*max+idx],norm[idx*max+i],b*max+i);
b++;
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}




for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
HX[idx] += HX[idx+ stride];
HY[idx] += HY[idx+ stride];
px[idx] += px[idx+ stride];
py[idx] += py[idx+ stride];
entropy[idx] += entropy[idx+ stride];
}
// synchronize within threadblock
__syncthreads();
}


if(idx>9000){
HXY[idx]=abs(norm[idx]*(log10f((px[0]*py[0]))));
//printf("tid %d %f %f %f %f \n",idx,HXY[idx],px[0],py[0],norm[idx]);
}

for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
HXY[idx] += HXY[idx+ stride];


}
// synchronize within threadblock
__syncthreads();
}

if (idx == 0){
if(HX[0]>HY[0]){
IMC[0]=(entropy[0]-HXY[0])/HX[0];
//printf("x%f %f %f %f px%f %f\n",abs(IMC[0]),entropy[0],HXY[0],HX[0],px[0],py[0]);
}
else{
IMC[0]=entropy[0]-HXY[0]/HY[0];
//printf("y%f %f %f %f\n",abs(IMC[0]),entropy[0],HXY[0],HY[0]);
}
printf("IMC %f\n",abs(IMC[0]));
}
}