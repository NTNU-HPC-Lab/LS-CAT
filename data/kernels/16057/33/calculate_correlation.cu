#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_correlation(float *norm,float *corelation,float *miu_x,float *miu_y,float *stdx,float *stdy,int *ikj,float *dif_variance,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
int tid=threadIdx.x;
int i;
for(i=0;i<max;i++){
if(idx>=i*max && idx<(i+1)*(max)){
miu_x[idx]=i*norm[idx];
//printf("%d,i %d  %f %f \n",idx,i,miu_x[idx],norm[idx]);
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}
int blok=0;
for(i=0;i<max;i++){
if(blok==i && idx<max){
miu_y[blok*max+idx]=i*norm[idx*max+i];
//printf("%d %d,i %d  %f %f %d \n",idx,idx,i,miu_y[idx],norm[idx*max+i],idx*max+i);
blok++;
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}
for(i=0;i<max;i++){
if(idx>=i*max && idx<(i+1)*(max)){
stdx[idx]=((i-miu_x[0])*(i-miu_x[0]))*norm[idx];
//printf("%d,i %d  %f %f \n",idx,i,miu_x[idx],norm[idx]);
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}
int batas=0;
for(i=0;i<max;i++){
// printf("%d",batas);
if(batas==i && idx<max){
stdy[batas*max+idx]=((i-miu_y[0])*(i-miu_y[0]))*norm[idx*max+i];
//printf("%d %d,i %d  %f %f %d \n",idx,idx,i,stdy[idx],norm[idx*max+i],idx*max+i);
batas++;
}

//printf("xx %d %f\n",idx*i+idx,miu_x[idx]);
}
if(idx==0){
for(i=0;i<max;i++){
for(int j=0;j<max;j++){
ikj[max*i+j]=i*j;
//printf("tid %d %d\n",max*i+j,ikj[max*i+j]);
}
}
}
if(idx<size){
corelation[idx]=((ikj[idx]*norm[idx]));
//printf("%d %d,i %d  %f %f \n",idx,idx,i,corelation[idx],norm[idx]);
}
for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
corelation[idx] += corelation[idx+ stride];
//printf("%d %f\n",idx,corelation[idx]);
}
// synchronize within threadblock
__syncthreads();
}
for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{

miu_x[idx] += miu_x[idx+ stride];
stdy[idx] += stdy[idx+ stride];
miu_y[idx] += miu_y[idx+ stride];
stdx[idx] += stdx[idx+ stride];
// corelation[idx] += corelation[idx+ stride];
//printf("%d %f\n",idx,miu_x[idx]);
}
// synchronize within threadblock
__syncthreads();
}
int k=0;
if(idx==0){
for(i=0;i<max;i++){
for(int j=0;j<max;j++){
k=abs(i-j);
dif_variance[k]=((k-((miu_x[0]+miu_y[0])/2))*(k-((miu_x[0]+miu_y[0])/2)))*norm[k];

if(k=i){
dif_variance[k]+=dif_variance[i];
//printf("%d %f %f %f \n",k,dif_variance[k],(k-((miu_x[0]+miu_y[0])/2)),norm[k]);

}
}
}

}

for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
dif_variance[idx] +=dif_variance[idx+stride];
}
// synchronize within threadblock
__syncthreads();
}
if (idx == 0){

printf("correlation %f\n",abs(corelation[0]-miu_x[0]*miu_y[0])/stdx[0]*stdy[0]);
printf("variance %f\n",stdx[0]);
printf("difference variance %f\n",dif_variance[0]);
}
}