#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_135(int *A,int *glcm, const int nx, const int ny,int max){
int ix = threadIdx.x + blockIdx.x* blockDim.x;
int iy = threadIdx.y + blockIdx.y* blockDim.y;
unsigned int idx =iy*nx+ix;
int i;
int k=0;
for(i=0;i<nx-1;i++){
if(blockIdx.x==i && idx >i*nx){
k=max*A[idx]+A[idx+(nx-1)];
atomicAdd(&glcm[k],1);
}
}
__syncthreads();
}