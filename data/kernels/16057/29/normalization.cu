#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalization(int *glcm,float *norm,int Max,int sum){
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * Max + ix;
__syncthreads();
if(idx<(Max+1)*(Max+1)){
norm[idx]=float(glcm[idx])/float(sum);
}
}