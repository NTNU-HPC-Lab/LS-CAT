#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div225(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx - 1 ; ++i){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
posisi = matrix[Index + 1] * nx + matrix[Index + nx];
atomicAdd(&newMatrix[posisi],1);
printf("Index : %d %d\n",Index + 1, Index + nx);
}
}
}