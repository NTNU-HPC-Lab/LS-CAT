#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_225(int *A,int *glcm, const int nx, const int ny,int max){
int ix = threadIdx.x + blockIdx.x* blockDim.x;
int iy = threadIdx.y + blockIdx.y* blockDim.y;
unsigned int idx =iy*nx+ix;
int i;
int k=0;
for(i=1;i<nx;i++){
if(blockIdx.x==i && idx >i*nx){
k=max*A[idx]+A[idx-(nx+1)];
atomicAdd(&glcm[k],1);
}
}
__syncthreads();
}