#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div45(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx - 1 ; i++){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
posisi = matrix[Index + nx] * nx + matrix[Index + 1];
atomicAdd(&newMatrix[posisi],1);
printf("Index : %d %d\n",Index + nx , Index + 1);
}
}
}