#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_entropy(float *norm,float *entropy,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
//printf("%d\n",idx);
int tid=threadIdx.x;
if(idx<size && norm[idx] !=0){
entropy[idx]=-(norm[idx]*log10f(norm[idx]));
//printf("%d f3 %f \n",idx,entropy[idx]);
__syncthreads();
}
for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
entropy[idx] += entropy[idx+ stride];
}
// synchronize within threadblock
__syncthreads();
}

if (idx == 0){

printf("entropy %f\n",entropy[0]);
}
}