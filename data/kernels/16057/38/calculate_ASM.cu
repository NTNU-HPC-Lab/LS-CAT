#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_ASM(float *norm,float *ASM,float *mulMatrix,int Max){
//printf("%d\n",max);
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

// int Index = iy * N + ix;

for (int k = 0; k < Max; k++) {
// Accumulate results for a single element
// c[row * N + col] += a[row * N + k] * b[k * N + col];
// printf("C[%d] = a[%d] * b[%d]\n",row * N + col,row * N + k, k * N + col);
atomicAdd(&mulMatrix[row * Max + col],norm[row * Max + k] * norm[k * Max + col]);
}
int Index = blockIdx.x * blockDim.x + threadIdx.x;

atomicAdd(&ASM[0],mulMatrix[Index]);

if (Index == 0){

printf("ASM %f\n",ASM[0]);
}
}