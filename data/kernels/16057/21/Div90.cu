#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div90(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx - 1 ; ++i){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
if(Index == 0 || Index % 2 == 0){
posisi = matrix[Index + nx] * nx + matrix[Index];
atomicAdd(&newMatrix[posisi],1);

posisi = matrix[Index + (nx + 1)] * nx + matrix[Index + 1];
atomicAdd(&newMatrix[posisi],1);
printf("Index : %d %d dan %d %d\n",Index + nx , Index, Index + (nx + 1),Index + 1);
}
}
}
}