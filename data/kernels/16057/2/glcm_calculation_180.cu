#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_180(int *A,int *glcm, const int nx, const int ny,int max){
//int iy = threadIdx.y + blockIdx.y* blockDim.y;
unsigned int idx =blockIdx.x*nx+threadIdx.x;
int i;
int k=0;
for(i=0;i<nx;i++){
if(idx>=i*nx && idx<((i+1) *nx)-1){
k=max*A[idx+1]+A[idx];
atomicAdd(&glcm[k],1);
}
}
}