#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_idm(float *norm,float *idm,int*dif,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
//printf("%d\n",idx);
int tid=threadIdx.x;
if(idx<size){
idm[idx]=((float(1)/(1+dif[idx]))*(norm[idx]));
//printf("%d  %f %f %f\n",idx,idm[idx],norm[idx],(float(1)/(1+dif[idx])));
__syncthreads();
}
for (int stride = 1; stride < blockDim.x; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
idm[idx] += idm[idx+ stride];
//printf("%d %f\n",idx,idm[idx]);
}
// synchronize within threadblock
__syncthreads();
}

if (idx == 0){

printf("idm %f\n",idm[0]);
}
}