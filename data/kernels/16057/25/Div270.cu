#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div270(int *matrix , int *newMatrix,int nx,int ny,int Max){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int Index = iy * nx + ix;
int posisi = 0;

for(int i = 0 ; i < nx - 1 ; ++i){
if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
if(Index == 0 || Index % 2 == 0){
posisi = matrix[Index] * nx + matrix[Index + nx];
atomicAdd(&newMatrix[posisi],1);

posisi = matrix[Index + 1] * nx + matrix[Index + (nx + 1)];
atomicAdd(&newMatrix[posisi],1);
printf("Index : %d %d dan %d %d\n",Index,Index + nx , Index + 1, Index + (nx + 1));
}
}
}
}