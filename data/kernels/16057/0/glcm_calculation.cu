#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation(int *A,int *glcm,float *glcmNorm, const int nx, const int ny,int maxx)
{

int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;



//unsigned int idr = iy * (maxx+1) + ix;


int k,l;
int p;


//Calculate GLCM
if(idx < nx*ny ){
for(k=0;k<=maxx;k++){
for(l=0;l<=maxx;l++){
if((A[idx]==k) && (A[idx+1]==l)){
p=((maxx+1)*k) +l;
glcm[p]+=1;
}
}
}
}


//Normalization
int sum;
sum = 0;
if(idx<(maxx+1)*(maxx+1)){
for(k=0;k<((maxx+1)*(maxx+1));k++){
sum+=glcm[k];
}
}
// if(ix<1){
//     printf("sum %d \n ",sum);
// }
if(idx<((maxx+1)*(maxx+1))){
glcmNorm[idx] = float(glcm[idx])/float(sum);
}

float sums;

if(ix<1){
for(k=0;k<((maxx+1)*(maxx+1));k++){
sums += glcmNorm[k];

}
}

float f1;

f1=0;
if(ix<1){
for(k=0;k<((maxx+1)*(maxx+1));k++){
f1 = f1 + glcmNorm[k];

}
}
//mat[offset] = sqrt(mat[offset]);

float f2 = 0;
if(ix<1){
for(k=0;k<((maxx+1)*(maxx+1));k++){
f2 = f2 + k*k*sums;

}
}

float f3;
f3 = sqrt(f1);


float f4;

if(ix<1){
for(k=0;k<((maxx+1)*(maxx+1));k++){
f4 += (glcmNorm[k] * log10f(glcmNorm[k]));

}
}

//float sum_average=0;


// float f5;
// if(ix<1){
//     for(k=0;k<((maxx+1)*(maxx+1));k++){
//         f2 = f2 + k*k*sums;

//     }
// }

// for (int j = 0, int i = 0; j<DIM, i<DIM; j++,i++){
//         for (int k = DIM*j; k<DIM*(j+1); k++)
//         f5 += i*mat[k];
//     }

// float f6;
// for (int i = 0; i<DIM; i++ ){
//     mat2[offset]= (i-f5)*(i-f5)*mat[offset];
//     for (int j=0; j<DIM; j++){
//         f6 += mat2[row*DIM*j];
//     }
// }
// if(row<DIM){
//         printf("array di device %d : %f \n",offset,mat[tidx]);
//         //printf("array di device %d : %f \n",offset,mat2[tidx]);
//         //mat[offset]=mat[offset]/sum;
// }
if(ix<1){
printf("ASM : %.1f\n", f1);
printf("Contrast : %.1f\n",f2);
printf("Energy : %.1f\n",f3);
printf("Entropy : %.1f\n",f4);
//printf("Miu : %.1f\n",f5);
//printf("Variance : %.1f\n",f6);
}

}