#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalization(int *glcm,float *norm,int max,int sum){
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
__syncthreads();
if(idx<(max+1)*(max+1)){
norm[idx]=float(glcm[idx])/float(sum);
}
}