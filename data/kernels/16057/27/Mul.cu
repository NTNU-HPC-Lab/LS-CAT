#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Mul(float *newMatrix,float *mulMatrix,int Max,float *sumMatrix){
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;

// int Index = iy * nx + ix;

for (int k = 0; k < Max; k++) {
// Accumulate results for a single element
// c[row * nx + col] += a[row * nx + k] * b[k * nx + col];
// printf("C[%d] = a[%d] * b[%d]\n",row * nx + col,row * nx + k, k * nx + col);
atomicAdd(&mulMatrix[row * Max + col],newMatrix[row * Max + k] * newMatrix[k * Max + col]);
// atomicAdd(&sumMatrix[0],mulMatrix[row * Max + col]);
}
}