#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_nol(int *A,int *glcm, const int nx, const int ny,int maxx)
{

int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;
//unsigned int idr = iy * (maxx+1) + ix;
int k,l;
int p;
//Calculate GLCM
if(idx < nx*ny ){
for(k=0;k<=maxx;k++){
for(l=0;l<=maxx;l++){
if((A[idx]==k) && (A[idx+1]==l)){
p=((maxx+1)*k) +l;
atomicAdd(&glcm[p],1);
}
}
}
}

}