#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_90(int *A,int *glcm, const int nx, const int ny,int max){
int ix = threadIdx.x + blockIdx.x* blockDim.x;
int iy = threadIdx.y + blockIdx.y* blockDim.y;
unsigned int idx =iy*nx+ix;
int i;
int k=0;
for(i=0;i<nx-1;i++){
if(idx>=i*nx && idx<((i+1) *nx)){
k=max*A[idx+nx]+A[idx];
atomicAdd(&glcm[k],1);
}
}
__syncthreads();
}