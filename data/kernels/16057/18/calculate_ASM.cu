#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_ASM(float *norm,float *ASM,int max,float sum,int size){
//printf("%d\n",max);
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * max + ix;
int tid=threadIdx.x;
if(idx<size){
ASM[idx]=norm[idx]*norm[idx];
// printf("%d asm %f\n",idx,norm[idx]);
}
//corelation[idx]=(((i*j)*norm[idx]));

for (int stride = 1; stride < size; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{

ASM[idx] += ASM[idx+stride];
//printf("%d %f %f\n",idx,corelation[idx],ASM[idx]);
}
// synchronize within threadblock
__syncthreads();
}

if (idx == 0){

printf("ASM %f %d\n",ASM[0],idx);
}
}