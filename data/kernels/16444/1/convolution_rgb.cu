#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_rgb(unsigned char *N,float *M,unsigned char* g,std::size_t cols, std::size_t rows,std::size_t mask_size){

int paddingSize = (( mask_size-1 )/2)*3;
unsigned int paddedH = cols + 2 * paddingSize;
unsigned int paddedW = rows*3 + 2 * paddingSize;

int i = blockIdx.x * blockDim.x + threadIdx.x + paddingSize ;
int j = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;
if( (i >= paddingSize) && (i < paddedW-paddingSize) && (j >= paddingSize) && (j<paddedH-paddingSize)) {
unsigned int oPixelPos = (i - paddingSize ) * cols + (j -paddingSize);
g[oPixelPos] = 0;
int iterationK = 0;
for(int k = -paddingSize; k <= paddingSize; k=k+3){
int iterationL = 0;
for(int l = -paddingSize; l<=paddingSize; l=l+3){
unsigned int iPixelPos = (i+k)*paddedH+(j+l);
unsigned int filtrePos = iterationK*mask_size + iterationL;

g[oPixelPos] += N[iPixelPos] * M[filtrePos];
iterationL++;
}
iterationK++;
}
}
}