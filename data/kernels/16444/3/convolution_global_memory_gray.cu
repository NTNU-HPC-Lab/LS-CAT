#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_global_memory_gray(unsigned char *N,float *M,unsigned char* g,std::size_t cols, std::size_t rows,std::size_t mask_size){
int paddingSize = ( mask_size-1 )/2;
unsigned int paddedH = cols + 2 * paddingSize;
unsigned int paddedW = rows + 2 * paddingSize;

int i = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
int j = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

if( (j >= paddingSize) && (j < paddedW-paddingSize) && (i >= paddingSize) && (i<paddedH-paddingSize)) {
unsigned int oPixelPos = (j - paddingSize ) * cols + (i -paddingSize);
for(int k = -paddingSize; k <= paddingSize; k++){
for(int l = -paddingSize; l<=paddingSize; l++){
unsigned int iPixelPos = (j+l)*cols+(i+k);
unsigned int coefPos = (k + paddingSize) * mask_size + (l+ paddingSize);
g[oPixelPos] += N[iPixelPos] * M[coefPos];
}
}
}
}