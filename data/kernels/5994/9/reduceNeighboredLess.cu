#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n){
// thread id
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// data pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;
// thread id out of range
if (threadIdx.x >= n) return;
for (int stride = 1; stride < blockDim.x; stride *= 2){
// first data index of this thread
int index = 2 * idx * stride;
// data add
if (index < blockDim.x){
idata[index] += idata[index + stride];
}
__syncthreads();
}
if (threadIdx.x == 0){
g_odata[blockIdx.x] = idata[0];
}
}