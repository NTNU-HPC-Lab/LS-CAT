#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n){
// thread id
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// data pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;
// thread id out of range
if (idx >= n) return;
for (int stride = 1; stride < blockDim.x; stride *= 2){
if (threadIdx.x % (stride*2) == 0){
idata[threadIdx.x] += idata[threadIdx.x + stride];
}
__syncthreads();
}
if (threadIdx.x == 0){
g_odata[blockIdx.x] = idata[0];
}
}