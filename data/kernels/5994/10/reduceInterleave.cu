#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleave(int *g_idata, int *g_odata, unsigned int n){
// thread id
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// data pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;
// thread id out of range
if (idx >= n) return;
for (int stride = blockDim.x/2; stride > 0; stride >>= 1){
if (threadIdx.x < stride){
idata[threadIdx.x] += idata[threadIdx.x + stride];
}
__syncthreads();
}
if (threadIdx.x == 0){
g_odata[blockIdx.x] = idata[0];
}
}