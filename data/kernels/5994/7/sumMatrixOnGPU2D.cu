#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int Nx, int Ny){
unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int idx = iy * Ny + ix;

if (ix < Nx && iy < Ny){
C[idx] = A[idx] + B[idx];
}
}