#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarp8(int *g_idata, int *g_odata, unsigned int n){
// thread id
int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;
// data pointer of this block(s)
int *idata = g_idata + blockIdx.x * blockDim.x * 8;
// unrolling blocks
if (idx + 7 * blockDim.x < n) {
int el0 = g_idata[idx];
int el1 = g_idata[idx + blockDim.x];
int el2 = g_idata[idx + 2*blockDim.x];
int el3 = g_idata[idx + 3*blockDim.x];
int el4 = g_idata[idx + 4*blockDim.x];
int el5 = g_idata[idx + 5*blockDim.x];
int el6 = g_idata[idx + 6*blockDim.x];
int el7 = g_idata[idx + 7*blockDim.x];
g_idata[idx] = el0+el1+el2+el3+el4+el5+el6+el7;
}
__syncthreads();

// unrolling in blocks
// 这种优化需要保证blockDim.x为2的k次幂，且最大为1024
if (blockDim.x >= 1024 && threadIdx.x < 512) idata[threadIdx.x] += idata[threadIdx.x + 512];
__syncthreads();
if (blockDim.x >= 512 && threadIdx.x < 256) idata[threadIdx.x] += idata[threadIdx.x + 256];
__syncthreads();
if (blockDim.x >= 256 && threadIdx.x < 128) idata[threadIdx.x] += idata[threadIdx.x + 128];
__syncthreads();
if (blockDim.x >= 128 && threadIdx.x < 64) idata[threadIdx.x] += idata[threadIdx.x + 64];
__syncthreads();

// unrolling sync in thread cluster(stride less than 32)
if (threadIdx.x < 32){
volatile int *vmem = idata;
vmem[threadIdx.x] += vmem[threadIdx.x + 32];
vmem[threadIdx.x] += vmem[threadIdx.x + 16];
vmem[threadIdx.x] += vmem[threadIdx.x + 8];
vmem[threadIdx.x] += vmem[threadIdx.x + 4];
vmem[threadIdx.x] += vmem[threadIdx.x + 2];
vmem[threadIdx.x] += vmem[threadIdx.x + 1];
}
if (threadIdx.x == 0){
g_odata[blockIdx.x] = idata[0];
}
}