#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned int n){
// thread id
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// data pointer of this block(s)
int *idata = g_idata + blockIdx.x * blockDim.x * 8;
// unrolling blocks
if (idx + 7 * blockDim.x < n) {
int el0 = g_idata[idx];
int el1 = g_idata[idx + blockDim.x];
int el2 = g_idata[idx + 2*blockDim.x];
int el3 = g_idata[idx + 3*blockDim.x];
int el4 = g_idata[idx + 4*blockDim.x];
int el5 = g_idata[idx + 5*blockDim.x];
int el6 = g_idata[idx + 6*blockDim.x];
int el7 = g_idata[idx + 7*blockDim.x];
g_idata[idx] = el0+el1+el2+el3+el4+el5+el6+el7;
}
__syncthreads();
// thread id out of range
if (idx >= n) return;
for (int stride = blockDim.x/2; stride > 32; stride >>= 1){
if (threadIdx.x < stride){
idata[threadIdx.x] += idata[threadIdx.x + stride];
}
__syncthreads();
}
// unrolling sync in blocks(stride less than 32)
if (threadIdx.x < 32){
volatile int *vmem = idata;
vmem[threadIdx.x] += vmem[threadIdx.x + 32];
vmem[threadIdx.x] += vmem[threadIdx.x + 16];
vmem[threadIdx.x] += vmem[threadIdx.x + 8];
vmem[threadIdx.x] += vmem[threadIdx.x + 4];
vmem[threadIdx.x] += vmem[threadIdx.x + 2];
vmem[threadIdx.x] += vmem[threadIdx.x + 1];
}
if (threadIdx.x == 0){
g_odata[blockIdx.x] = idata[0];
}
}