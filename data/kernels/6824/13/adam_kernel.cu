#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;

x[index] = x[index] + (rate * sqrtf(1.f-powf(B2, t)) / (1.f-powf(B1, t)) * m[index] / (sqrtf(v[index]) + eps));
}