#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add (float *d_A, float *d_B, float *d_C, int widthA, int widthB, int widthC)
{
int startA = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthA;
int startB = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthB;
int startC = blockIdx.x*64 + threadIdx.x*2 + (blockIdx.y*8 + threadIdx.y)*widthC;
float2 tempA = *(float2 *)(d_A+startA);
float2 tempB = *(float2 *)(d_B+startB);
tempA.x += tempB.x;
tempA.y += tempB.y;
*(float2 *)(d_C+startC) = tempA;
}