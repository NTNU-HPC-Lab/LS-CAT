#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kmeans_kernel(const sequence_t *data, const sequence_t *centroids, int * membership, unsigned int * tmp_centroidCount, unsigned int n, unsigned int numClusters )
{

int index = blockIdx.x * blockDim.x  + threadIdx.x;
if (index < n){

unsigned int min_distance = UINT_MAX;
long nearest = -1;

for (int i = 0; i < numClusters; i++){
sequence_t centroid = centroids[i];
unsigned int distance = __popcll(centroid.x ^ data[index].x) +
__popcll(centroid.y ^ data[index].y) +
__popcll(centroid.z ^ data[index].z);
if(distance < min_distance) {
nearest = i;
min_distance = distance;
}
}

if(membership[index] != nearest) {
membership[index]=nearest;
atomicInc(&cuda_delta,n*2);
}

unsigned int *tmp_centroid = &tmp_centroidCount[membership[index] * BIT_SIZE_OF(sequence_t)];
for (unsigned int j=0;j<SEQ_DIM_BITS_SIZE;j++)  {
// bits tmp_centroid[0] is less significative bit from sequence_t
// bits tmp_centroid[0] = z << 0
unsigned long int mask = 1;
if (data[index].z & (mask << j)){
atomicInc(&tmp_centroid[j],n);
}
if (data[index].y & (mask << j)){
atomicInc(&tmp_centroid[SEQ_DIM_BITS_SIZE + j],n);
}
if (data[index].x & (mask << j)){
atomicInc(&tmp_centroid[(2 *SEQ_DIM_BITS_SIZE)+j],n);
}
}
}
}