#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int cuda_delta = 0;  __device__ unsigned int maskForMode(unsigned int x, unsigned int y, unsigned int z, unsigned int w ){
unsigned int max = x > y ? x : y;
max = z > max ? z : max;
max = w > max ? w : max;
unsigned int mask = 0;

if (max == x){
mask |= 1;
}
if (max == y){
mask |= 2; // 010
}
if (max == z){
mask |= 4; // 0100
}
if (max == w){
mask |= 8; // 0100
}
return mask;
}
__global__ void update_centroids(const sequence_t *data, sequence_t *centroids, unsigned int * tmp_centroidCount, unsigned int numClusters){
int i = blockIdx.x * blockDim.x  + threadIdx.x;

if (i < numClusters){
sequence_t seq = make_ulong3(0,0,0);
unsigned int *tmp_centroid = &tmp_centroidCount[i* BIT_SIZE_OF(sequence_t)];
for (int j=0;j<SEQ_DIM_BITS_SIZE;j+=4)
{

// bits tmp_centroid[0] is less significative bit from sequence_t
// bits tmp_centroid[0] = z << 0
unsigned int *bitCountX = &tmp_centroid[j + (SEQ_DIM_BITS_SIZE * 2)];
unsigned int *bitCountY = &tmp_centroid[j + SEQ_DIM_BITS_SIZE];
unsigned int *bitCountZ = &tmp_centroid[j];

unsigned long int mask = maskForMode(bitCountX[0],bitCountX[1],bitCountX[2],bitCountX[3]);
seq.x |= (mask << j);
mask = maskForMode(bitCountY[0],bitCountY[1],bitCountY[2],bitCountY[3]);
seq.y |= (mask << j);
mask = maskForMode(bitCountZ[0],bitCountZ[1],bitCountZ[2],bitCountZ[3]);
seq.z |= (mask << j);
}
centroids[i] = seq;
}
}