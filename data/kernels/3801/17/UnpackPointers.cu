#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
const int tx = threadIdx.x;
int numPts = 0;
for (int y=0;y<h/32;y++) {
for (int x=0;x<w;x+=16) {
unsigned int val = minmax[y*w+x+tx];
if (val) {
for (int k=0;k<32;k++) {
if (val&0x1 && numPts<maxPts) {
ptrs[16*numPts+tx] = (y*32+k)*w + x+tx;
numPts++;
}
val >>= 1;
}
}
}
}
}