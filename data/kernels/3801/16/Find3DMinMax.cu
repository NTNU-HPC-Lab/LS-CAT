#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
// Data cache
__shared__ float data1[3*(MINMAX_W + 2)];
__shared__ float data2[3*(MINMAX_W + 2)];
__shared__ float data3[3*(MINMAX_W + 2)];
__shared__ float ymin1[(MINMAX_W + 2)];
__shared__ float ymin2[(MINMAX_W + 2)];
__shared__ float ymin3[(MINMAX_W + 2)];
__shared__ float ymax1[(MINMAX_W + 2)];
__shared__ float ymax2[(MINMAX_W + 2)];
__shared__ float ymax3[(MINMAX_W + 2)];

// Current tile and apron limits, relative to row start
const int tx = threadIdx.x;
const int xStart = blockIdx.x*MINMAX_W;
const int xEnd = xStart + MINMAX_W - 1;
const int xReadPos = xStart + tx - WARP_SIZE;
const int xWritePos = xStart + tx;
const int xEndClamped = min(xEnd, width - 1);
int memWid = MINMAX_W + 2;

int memPos0 = (tx - WARP_SIZE + 1);
int memPos1 = (tx - WARP_SIZE + 1);
int yq = 0;
unsigned int output = 0;
for (int y=0;y<32+2;y++) {

output >>= 1;
int memPos =  yq*memWid + (tx - WARP_SIZE + 1);
int yp = 32*blockIdx.y + y - 1;
yp = max(yp, 0);
yp = min(yp, height-1);
int readStart = yp*pitch;

// Set the entire data cache contents
if (tx>=(WARP_SIZE-1)) {
if (xReadPos<0) {
data1[memPos] = 0;
data2[memPos] = 0;
data3[memPos] = 0;
} else if (xReadPos>=width) {
data1[memPos] = 0;
data2[memPos] = 0;
data3[memPos] = 0;
} else {
data1[memPos] = d_Data1[readStart + xReadPos];
data2[memPos] = d_Data2[readStart + xReadPos];
data3[memPos] = d_Data3[readStart + xReadPos];
}
}
__syncthreads();

int memPos2 = yq*memWid + tx;
if (y>1) {
if (tx<memWid) {
float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
ymin1[tx] = min1;
ymin2[tx] = fminf(fminf(min1, min2), min3);
ymin3[tx] = min3;
ymax1[tx] = max1;
ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
ymax3[tx] = max3;
}
}
__syncthreads();

if (y>1) {
if (tx<MINMAX_W) {
if (xWritePos<=xEndClamped) {
float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
minv = fminf(minv, d_Threshold[1]);
float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
maxv = fmaxf(maxv, d_Threshold[0]);
if (data2[memPos1+1]<minv || data2[memPos1+1]>maxv)
output |= 0x80000000;
}
}
}
__syncthreads();

memPos0 = memPos1;
memPos1 = memPos2;
yq = (yq<2 ? yq+1 : 0);
}
if (tx<MINMAX_W && xWritePos<width) {
int writeStart = blockIdx.y*pitch + xWritePos;
d_Result[writeStart] = output;
}
}