#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
const int x = blockIdx.x*SUBTRACTM_W + threadIdx.x;
const int y = blockIdx.y*SUBTRACTM_H + threadIdx.y;
int sz = height*pitch;
int p = threadIdx.z*sz + y*pitch + x;
if (x<width && y<height)
d_Result[p] = d_Data[p] - d_Data[p + sz];
__syncthreads();
}