#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;




__global__ void sumCols(int* d_matrix, int* d_result, int numRows, int numCols) {
int sum = 0;

int idx = blockDim.x * blockIdx.x + threadIdx.x;

for (int i = 0; i < numRows; i++) {
sum += d_matrix[idx + (numCols * i)];
}

d_result[idx] = sum;
}