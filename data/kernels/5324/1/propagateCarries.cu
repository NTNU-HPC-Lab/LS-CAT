#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;




__global__ void propagateCarries(int* d_matrix, int numCols) {
int idx = blockDim.x * blockIdx.x + threadIdx.x * numCols;
int carry = 0;

for (int i = numCols - 1; i >= 0; i--) {
int rowVal = (d_matrix[idx + i] + carry) % 10;
carry = (d_matrix[idx + i] + carry) / 10;

d_matrix[idx + i] = rowVal;
}
}