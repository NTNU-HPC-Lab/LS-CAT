#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;




__global__ void multiplyDigits(char* d_str1, char* d_str2, int* d_matrix, int str1_len, int str2_len) {
int row = blockDim.y * blockIdx.x + threadIdx.y;
int col = blockDim.x * blockIdx.y + threadIdx.x;

int idx = row * str1_len + (col + (str2_len * row)) + 1 + (row);

d_matrix[idx] = (d_str2[row] - '0') * (d_str1[col] - '0');
}