#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* Copyright 2010 Marco Hutter - http://www.jcuda.org
*/


/**
* Kernels for the JCudaDriverTextureTest class. These
* kernels will read data via the texture references at
* the given positions, and store the value that is
* read into the given output memory.
*/

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"


extern "C"






__global__ void test_float4_2D(float4 *output, float posX, float posY)
{
float4 result = tex2D(texture_float4_2D, posX, posY);
output[0] = result;
}