#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* Copyright 2010 Marco Hutter - http://www.jcuda.org
*/


/**
* Kernels for the JCudaDriverTextureTest class. These
* kernels will read data via the texture references at
* the given positions, and store the value that is
* read into the given output memory.
*/

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"


extern "C"






__global__ void test_float4_1D(float4 *output, float posX)
{
float4 result = tex1D(texture_float4_1D, posX);
output[0] = result;
}