#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeKernelPhase(float* KernelPhase, int row, int column, float* ImgProperties) {

const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
const float pixdxInv = ImgProperties[1] / ImgProperties[0]; // Magnification/pixSize
const float km = ImgProperties[2] / ImgProperties[3]; // nm / lambda


for (int i = threadID; i < row*column; i += numThreads) {
int dx = i % row;
int dy = i / row;

dx = ((dx - row / 2)>0) ? (dx - row) : dx;
dy = ((dy - row / 2)>0) ? (dy - row) : dy;

float kdx = float(dx)*pixdxInv/row;
float kdy = float(dy)*pixdxInv/row;
float temp = km*km - kdx*kdx - kdy*kdy;
KernelPhase[i] = (temp >= 0) ? (sqrtf(temp)-km) : 0;
}


}