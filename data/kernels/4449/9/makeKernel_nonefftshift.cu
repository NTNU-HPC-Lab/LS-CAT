#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeKernel_nonefftshift(float* KernelPhase, int row, int column, float* ImgProperties) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
float pixSize = ImgProperties[0];
float MagX = ImgProperties[1];
float nmed = ImgProperties[2];
float lambda = ImgProperties[3];
float MagXscaling = 1/ImgProperties[4];
float pixdxInv = MagX / pixSize*MagXscaling; // Magnification/pixSize
float km = nmed / lambda; // nmed / lambda


for (int i = threadID; i < row*column; i += numThreads) {
int dx = i % row;
int dy = i / row;

dx= ((dx - row / 2)>0) ? (dx - row) : dx;
dy= ((dy - row / 2)>0) ? (dy - row) : dy;

float kdx = float(dx)*pixdxInv;
float kdy = float(dy)*pixdxInv;
float temp = km*km - kdx*kdx - kdy*kdy;
KernelPhase[i] = (temp >= 0) ? (sqrtf(temp)-km) : 0;
}
}