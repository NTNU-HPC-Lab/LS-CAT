#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeKernel(float* KernelPhase, int row, int column, float* ImgProperties, float MagXscaling) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
float MagX = ImgProperties[1];
float pixSize= ImgProperties[0];
float nm = ImgProperties[2];
float lambda = ImgProperties[3];


float pixdxInv = MagX/pixSize*MagXscaling; // Magnification/pixSize
float km = nm/lambda; // nm / lambda

for (int i = threadID; i < row*column; i += numThreads) {
int dx = i%row;
int dy = i/row;

float kdx = float( dx - row/2)*pixdxInv;
float kdy = float( dy - row/2)*pixdxInv;
float temp = km*km - kdx*kdx - kdy*kdy;
KernelPhase[i]= (temp >= 0) ? (sqrtf(temp)-km) : 0;


//This still needs quadrant swapping so this will not work in the ifft routine as is!



}
}