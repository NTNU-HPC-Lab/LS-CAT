#include "hip/hip_runtime.h"
#include "includes.h"
// customDllFunctions.cu

//////////////////////////
// Template to write .dlls
//////////////////////////

/* Include the following directories for the program to run appropriately:
///////////////////////
in the VC++ directories:

$(VC_IncludePath);
$(WindowsSDK_IncludePath);
C:\ProgramData\NVIDIA Corporation\CUDA Samples\v9.0\common\inc;
$(CUDA_INC_PATH)
C:\Program Files\National Instruments\LabVIEW 2015\cintools

////////////////////////
CUDA/C/C++ directories:
./
../../common/inc
$(CudaToolkitDir)/include

////////////////////////////////
Linker/General include libraries:
cudart.lib

//changed the target machine platform from 32 to 64 bit
*/





////////////////////////////////////////////////////////////////////////////////
// Complex operations,
////////////////////////////////////////////////////////////////////////////////


__global__ void ScaleArray(float *d_a, float alpha, int arraySize)
{
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
float temp;

for (int i = threadID; i < arraySize; i += numThreads)
{
temp = d_a[i];
d_a[i] = alpha*temp;
}
}