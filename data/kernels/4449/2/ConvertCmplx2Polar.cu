#include "hip/hip_runtime.h"
#include "includes.h"
// customDllFunctions.cu

//////////////////////////
// Template to write .dlls
//////////////////////////

/* Include the following directories for the program to run appropriately:
///////////////////////
in the VC++ directories:

$(VC_IncludePath);
$(WindowsSDK_IncludePath);
C:\ProgramData\NVIDIA Corporation\CUDA Samples\v9.0\common\inc;
$(CUDA_INC_PATH)
C:\Program Files\National Instruments\LabVIEW 2015\cintools

////////////////////////
CUDA/C/C++ directories:
./
../../common/inc
$(CudaToolkitDir)/include

////////////////////////////////
Linker/General include libraries:
cudart.lib

//changed the target machine platform from 32 to 64 bit
*/





////////////////////////////////////////////////////////////////////////////////
// Complex operations,
////////////////////////////////////////////////////////////////////////////////


__device__ static __inline__ float cmagf(float x, float y)
{
float a, b, v, w, t;
a = fabsf(x);
b = fabsf(y);
if (a > b) {
v = a;
w = b;
}
else {
v = b;
w = a;
}
t = w / v;
t = 1.0f + t * t;
t = v * sqrtf(t);
if ((v == 0.0f) || (v > 3.402823466e38f) || (w > 3.402823466e38f)) {
t = v + w;
}
return t;
}
__global__ void ConvertCmplx2Polar(float* inRe, float* inIm, float* mag, float* phase, int size) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
for (int i = threadID; i < size; i += numThreads)
{
phase[i] = atan2f(inIm[i], inRe[i]);
mag[i] = cmagf(inIm[i], inRe[i]);
}
}