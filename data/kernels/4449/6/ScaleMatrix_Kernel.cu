#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ScaleMatrix_Kernel(float *d_a, float alpha, int arraySize)
{
// Block index
int bx = blockIdx.x;

// Thread index
int tx = threadIdx.x;
int begin = blockDim.x * bx;
int index = begin + tx;

// copies array into shared memory, important only if threads are communicating between each other. Its not necessary here since we are only scaling vector.

__shared__ float d_as[BLOCKSIZE];

d_as[tx] = d_a[index];

__syncthreads();

// copies array back to global device memory

d_a[index] = alpha * d_as[tx];

}