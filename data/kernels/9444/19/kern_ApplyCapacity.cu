#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ApplyCapacity(float* sinkBuffer, float* capBuffer, int size)
{
int idx = CUDASTDOFFSET;
float value = sinkBuffer[idx];
float cap = capBuffer[idx];
value = (value < 0.0f) ? 0.0f: value;
value = (value > cap) ? cap: value;
if( idx < size )
{
sinkBuffer[idx] = value;
}
}