#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_UpdateLabel(float* sinkBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float CC, int size)
{
int idx = CUDASTDOFFSET;
float value = labelBuffer[idx] + CC*(incBuffer[idx] - divBuffer[idx] - sinkBuffer[idx]);
value = saturate(value);
if( idx < size )
{
labelBuffer[idx] = value;
}
}