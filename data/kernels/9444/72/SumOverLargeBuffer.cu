#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumOverLargeBuffer( float* buffer, int spread, int size ){

int offset = CUDASTDOFFSET;
float value1 = buffer[offset];
float value2 = buffer[offset+spread];

if( offset+spread < size )
buffer[offset] = value1+value2;

}