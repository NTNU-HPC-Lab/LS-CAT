#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ResetSinkBuffer(float* sink, float* source, float* div, float* label, float ik, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = (1.0f-ik)*sink[idx] + ik*(source[idx] - div[idx] + label[idx] * iCC);
if( idx < size )
{
sink[idx] = value;
}
}