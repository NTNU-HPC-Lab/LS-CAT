#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_FindLeafSinkPotential(float* sinkBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = incBuffer[idx] - divBuffer[idx] + labelBuffer[idx] * iCC;
if( idx < size )
{
sinkBuffer[idx] = value;
}
}