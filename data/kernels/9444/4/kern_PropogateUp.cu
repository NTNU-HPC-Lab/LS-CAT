#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_PropogateUp(float* working, int span, int imageSize)
{
int idx = CUDASTDOFFSET;
float inputValue1 = working[idx];
float inputValue2 = working[idx+span];
float outputVal = (inputValue1 > inputValue2) ? inputValue1: inputValue2;
if(idx+span < imageSize)
{
working[idx] = outputVal;
}
}