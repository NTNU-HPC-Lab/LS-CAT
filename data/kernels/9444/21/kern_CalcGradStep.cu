#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_CalcGradStep(float* sinkBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float stepSize, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = stepSize*(sinkBuffer[idx] + divBuffer[idx] - incBuffer[idx] - labelBuffer[idx] * iCC);
if( idx < size )
{
divBuffer[idx] = value;
}
}