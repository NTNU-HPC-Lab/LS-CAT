#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ConvertBuffer(short* agreement, float* output, int size )
{
int idx = CUDASTDOFFSET;
float locAgreement = (float) agreement[idx];
if( idx < size )
{
output[idx] = locAgreement;
}
}