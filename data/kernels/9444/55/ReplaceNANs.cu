#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReplaceNANs(float* buffer, float value, int size){
int offset = CUDASTDOFFSET;
float current = buffer[offset];
current = isfinite(current) ? current : value;
if(offset < size ) buffer[offset] = current;
}