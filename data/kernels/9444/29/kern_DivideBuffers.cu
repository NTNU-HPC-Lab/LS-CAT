#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_DivideBuffers(float* dst, float* src, const int size)
{
int idx = CUDASTDOFFSET;
float value1 = src[idx];
float value2 = dst[idx];
float minVal =  value2 / value1;
if( idx < size )
{
dst[idx] = minVal;
}
}