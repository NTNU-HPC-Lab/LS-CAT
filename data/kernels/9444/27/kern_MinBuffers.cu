#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_MinBuffers(float* b1, float* b2, int size)
{
int idx = CUDASTDOFFSET;
float value1 = b1[idx];
float value2 = b2[idx];
float minVal =  (value1 < value2) ? value1 : value2;
if( idx < size )
{
b1[idx] = minVal;
}
}