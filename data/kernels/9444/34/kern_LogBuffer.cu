#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_LogBuffer(float* agreement, float* output, float maxOut, int size, short max)
{
int idx = CUDASTDOFFSET;
float locAgreement = (float) agreement[idx];
float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
logValue = (logValue > 0.0f) ? logValue : 0.0f;
logValue = (logValue < maxOut) ? logValue: maxOut;
if( idx < size )
{
output[idx] = logValue;
}
}