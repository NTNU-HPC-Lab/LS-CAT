#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ProbBuffer(float* agreement, float* output, int size, short max)
{
int idx = CUDASTDOFFSET;
float locAgreement = agreement[idx];
float probValue = (float) locAgreement / (float) max;
probValue = (probValue < 1.0f) ? probValue: 1.0f;
if( idx < size )
{
output[idx] = probValue;
}
}