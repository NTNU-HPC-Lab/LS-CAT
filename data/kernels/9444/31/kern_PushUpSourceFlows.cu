#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_PushUpSourceFlows(float* psink, float* sink, float* source, float* div, float* label, float w, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = psink[idx] + w*(sink[idx] - source[idx] + div[idx] - label[idx] * iCC);
if( idx < size )
{
psink[idx] = value;
}
}