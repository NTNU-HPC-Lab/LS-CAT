#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_FindSinkPotentialAndStore(float* workingBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = workingBuffer[idx] + incBuffer[idx] - divBuffer[idx] + labelBuffer[idx] * iCC;
if( idx < size )
{
workingBuffer[idx] = value;
}
}