#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CUDAkernel_multiply( float* sourceA, float* sourceB, float* destination, int size )
{
int index = CUDASTDOFFSET;
float a = sourceA[index];
float b = sourceB[index];
if( index < size )
{
destination[index] = a * b;
}
}