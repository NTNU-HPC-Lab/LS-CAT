#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_BlurBuffer(float* input, float* output, int size, int spread, int dim)
{
int idx = CUDASTDOFFSET;
int x = (idx / spread) % dim;
float curr = input[idx];
float down = (idx-spread >= 0)   ? input[idx-spread] : 0;
float up   = (idx+spread < size) ? input[idx+spread] : 0;
float newVal = 0.7865707f * curr + 0.1064508f * ((x > 0 ? down : curr) + (x < dim-1 ? up : curr));
__syncthreads();
if( idx < size )
{
output[idx] = newVal;
}
}