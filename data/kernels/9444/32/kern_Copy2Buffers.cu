#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_Copy2Buffers(float* fIn, float* fOut1, float* fOut2, int size)
{
int idx = CUDASTDOFFSET;
float value = fIn[idx];
if( idx < size )
{
fOut1[idx] = value;
}
if( idx < size )
{
fOut2[idx] = value;
}
}