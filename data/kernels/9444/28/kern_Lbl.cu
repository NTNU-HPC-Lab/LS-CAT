#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_Lbl(float* lbl, float* flo, float* cap, const int size)
{
int idx = CUDASTDOFFSET;
float value1 = cap[idx];
float value2 = flo[idx];
float minVal =  (value2 == value1) ? 1.0f : 0.0f;
if( idx < size )
{
lbl[idx] = minVal;
}
}