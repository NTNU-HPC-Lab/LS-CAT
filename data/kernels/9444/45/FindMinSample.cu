#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinSample(float* DistanceBuffer, short2* IndexBuffer, int spread, int mapSizeX, int mapSizeY)
{
int kOffset = CUDASTDOFFSET;
float distance1 = DistanceBuffer[kOffset];
float distance2 = DistanceBuffer[kOffset + spread];
short2 index1 = IndexBuffer[kOffset];
short2 index2 = IndexBuffer[kOffset + spread];

if (kOffset + spread < mapSizeX * mapSizeY)
{
DistanceBuffer[kOffset] = (distance1 < distance2) ? distance1 : distance2;
IndexBuffer[kOffset] = (distance1 < distance2) ? index1 : index2;
}
}