#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_NormLogBuffer(float* agreement, float* output, float maxOut, int size, short max)
{
int idx = CUDASTDOFFSET;
float locAgreement = (float) agreement[idx];
float logValue = (locAgreement > 0.0f) ? log((float)max)-log(locAgreement): maxOut;
logValue = (logValue > 0.0f) ? logValue : 0.0f;
logValue = (logValue < maxOut) ? logValue / maxOut: 1.0f;
if( idx < size )
{
output[idx] = logValue;
}
}