#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CUDAkernel_accumulate( float* buffer, int addSize, int size )
{
int index = CUDASTDOFFSET;
float a = buffer[index];
float b = buffer[index+addSize];
if( index+addSize < size )
{
buffer[index] = a+b;
}
}