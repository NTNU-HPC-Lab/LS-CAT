#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_FindSourcePotentialAndStore(float* workingBuffer, float* sinkBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
int idx = CUDASTDOFFSET;
float value = workingBuffer[idx] + sinkBuffer[idx] + divBuffer[idx] - labelBuffer[idx] * iCC;
if( idx < size )
{
workingBuffer[idx] = value;
}
}