#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "testKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_matrixA = NULL;
hipMalloc(&d_matrixA, XSIZE*YSIZE);
float *d_matrixB = NULL;
hipMalloc(&d_matrixB, XSIZE*YSIZE);
float *d_matrixC = NULL;
hipMalloc(&d_matrixC, XSIZE*YSIZE);
const unsigned int ah = 1;
const unsigned int aw = 1;
const unsigned int bh = 1;
const unsigned int bw = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
testKernel<<<gridBlock,threadBlock>>>(d_matrixA,d_matrixB,d_matrixC,ah,aw,bh,bw);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
testKernel<<<gridBlock,threadBlock>>>(d_matrixA,d_matrixB,d_matrixC,ah,aw,bh,bw);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
testKernel<<<gridBlock,threadBlock>>>(d_matrixA,d_matrixB,d_matrixC,ah,aw,bh,bw);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}