#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

/* matrix project which demonstrates the basics on how to setup a project
* example application.
* Device code.
*/

#ifndef _matrix_KERNEL_H_
#define _matrix_KERNEL_H_





#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////

#endif // #ifndef _matrix_KERNEL_H_
__global__ void testKernel(	float* d_matrixA, float* d_matrixB, float* d_matrixC, const unsigned int ah, const unsigned int aw, const unsigned int bh, const unsigned int bw) {
// shared memory - Matrix B
#ifdef CHANGE4
__shared__ float shm_matrixB[KERNEL_SIZE+(2*KERNEL_LENGTH)];
#elif defined(CHANGE1)
__shared__ float shm_matrixB[KERNEL_SIZE];
#endif

// shared memory - SubMatrix A
#ifdef CHANGE4
__shared__ float shm_subMatrixA[BLOCK_SIZE_HEIGHT*BLOCK_SIZE_WIDTH+WARP_SIZE];

#elif defined(CHANGE3)
__shared__ float shm_subMatrixA0[BLOCK_SIZE_HEIGHT*BLOCK_SIZE_WIDTH];
__shared__ float shm_subMatrixA1[BLOCK_SIZE_HEIGHT*BLOCK_SIZE_WIDTH];

#elif defined(CHANGE2)
__shared__ float shm_subMatrixA[2*BLOCK_SIZE_HEIGHT*BLOCK_SIZE_WIDTH];

#endif

// the size is determined by the host application
const unsigned int bx = blockIdx.x;
const unsigned int by = blockIdx.y;


// access thread id
const int tx = threadIdx.x;
const int ty = threadIdx.y;

#ifdef CHANGE3
int xstep = bx;
int ystep = 2 * by;
#elif defined(CHANGE2)
int xstep = bx;
int ystep = by;
#else
int xstep = BLOCK_SIZE * bx;
int ystep = BLOCK_SIZE * by;
#endif

#ifdef CHANGE3
float sum0 = 0;
float sum1 = 0;

#else
float sum = 0;
#endif

int y = ystep + ty;
int x = xstep + tx;


#ifdef CHANGE4

if(tx<(KERNEL_LENGTH))
{// Padding zeros to get rid of dependence on divergence
shm_matrixB[ tx ] = 0;
shm_matrixB[ KERNEL_SIZE + tx ] = 0;
}

// Padding zeros to get rid of dependence on divergence
if(tx<(KERNEL_SIZE))
shm_matrixB[ tx + KERNEL_LENGTH ] = d_matrixB[ tx ];

if(tx<(WARP_SIZE))
shm_subMatrixA[ tx  ] = 0;

__syncthreads();


#elif defined(CHANGE1)

if((tx<(KERNEL_SIZE)))
shm_matrixB[ tx ] = d_matrixB[ tx ];
//	__syncthreads();


#endif


/* -------------------------------- Computation -------------------------------------*/

#ifdef CHANGE4
//modified code
for (int j=0; j<bh+1; j++) {


shm_subMatrixA[tx+WARP_SIZE] = 0;

if ((y-j+1)>-1)
{
shm_subMatrixA[tx+WARP_SIZE] = d_matrixA[(y-j+1)*aw+(x)];
}

__syncthreads();

for(int k = 0; k < bw; ++k) {
float b0 = shm_matrixB[j*bw+k];
float b1 = shm_matrixB[(j+1)*bw+k];
float a = 0;

a = shm_subMatrixA[tx-k+WARP_SIZE];
sum0 += a*b0;
sum1 += a*b1;
}//k loop
__syncthreads();
}//j loop


#elif defined(CHANGE3)
//modified code
for (int j=0; j<bh; j++) {

if ((((y-j)>-1) &&(y-j)<ah))
{
shm_subMatrixA0[tx] = d_matrixA[(y-j)*aw+(x)];
}
if ((((y+1-j)>-1) &&(y+1-j)<ah))
{
shm_subMatrixA1[tx] = d_matrixA[(y+1-j)*aw+(x)];
}

__syncthreads();

for(int k = 0; k < bw; ++k) {
float b = shm_matrixB[j*bw+k];
float a0 = 0;
float a1 = 0;
// check the out-of-bound
if ((((y-j)>-1) &&(y-j)<ah)&&(x-k)>-1&&(x-k)<aw) {

a0 = shm_subMatrixA0[tx-k];

sum0 += a0*b;
}
if ((((y+1-j)>-1) &&(y+1-j)<ah)&&(x-k)>-1&&(x-k)<aw) {

a1 = shm_subMatrixA1[tx-k];

sum1 += a1*b;
}
}//k loop
__syncthreads();
}//j loop


#elif defined(CHANGE2)
//modified code
for (int j=0; j<bh; j++) {

#if 0
if(tx<WARP_SIZE)
if (((y-j)>-1) &&((y-j)<ah)&&((x-DATA_TO_PULL_SIZE)>-1)&&((x - DATA_TO_PULL_SIZE)<aw))
shm_subMatrixA[tx] = d_matrixA[(y-j)*aw+(x-DATA_TO_PULL_SIZE)];
#endif

if ((((y-j)>-1) &&(y-j)<ah))
shm_subMatrixA[tx] = d_matrixA[(y-j)*aw+(x)];

__syncthreads();

for(int k = 0; k < bw; ++k) {
float b = shm_matrixB[j*bw+k];
float a = 0;
// check the out-of-bound
if ((y-j)>-1 &&(y-j)<ah&&((x)-k)>-1&&((x)-k)<aw) {
a = shm_subMatrixA[tx-k];

sum += a*b;
}
}//k loop
__syncthreads();
}//j loop
#elif defined(CHANGE1)
//modified code
for (int j=0; j<bh; j++) {
for(int k = 0; k < bw; ++k) {
float b = shm_matrixB[j*bw+k];
float a = 0;
// check the out-of-bound
if ((y-j)>-1&&(y-j)<ah&&(x-k)>-1&&(x-k)<aw) {
a = d_matrixA[(y-j)*aw+(x-k)];
sum += a*b;
}
}
} //j loop
__syncthreads();
#else
//Original Code
for (int j=0; j<bh; j++) {
for(int k = 0; k < bw; ++k) {
float b = d_matrixB[j*bw+k];
float a = 0;
// check the out-of-bound
if ((y-j)>-1&&(y-j)<ah&&(x-k)>-1&&(x-k)<aw) {
a = d_matrixA[(y-j)*aw+(x-k)];
sum += a*b;
}
}
}//j loop
#endif //CHANGES


#ifdef CHANGE4
// write data to global memory
d_matrixC[(1*y*aw)+x] = sum0;
d_matrixC[(((1*y)+1)*aw)+x] = sum1;
#elif defined(CHANGE3)
// write data to global memory
d_matrixC[(1*y*aw)+x] = sum0;
d_matrixC[(((1*y)+1)*aw)+x] = sum1;

#else
// write data to global memory
d_matrixC[y*aw+x] = sum;
#endif
}// end of func