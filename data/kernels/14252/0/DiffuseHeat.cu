#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDA_INPUT "input.txt"
#define CUDA_OUTPUT "cuda_output.txt"

int NUMPOINTS;
double ENDTIME;
double DT;
double ENDVALUES;

void InitialiseToZero(float* array);
__device__ void PrintPointsGPU(float* array, int size, double currentTime);
void PrintPointsCPU(float* array, double currentTime);
void ProcessOutput(float* array, int testCase, float time);
void CheckPoints(float* firstArray, float* secondArray);


__global__ void DiffuseHeat(float* currentPoints, float* nextPoints, const size_t size, double dx, double dt, const size_t endTime)
{
unsigned int threadIndex = (threadIdx.x + blockDim.x * blockIdx.x) + 1;
double currentTime = 0.0;
if (threadIndex > 0 && threadIndex < size-1)
{
while (currentTime < endTime)
{
nextPoints[threadIndex] = currentPoints[threadIndex] + 0.25*(currentPoints[threadIndex+1] - (2*currentPoints[threadIndex]) + currentPoints[threadIndex-1]);
__syncthreads();
currentPoints[threadIndex] = nextPoints[threadIndex];
currentTime += dt;
__syncthreads();
}
}
}