#include "hip/hip_runtime.h"
#include "includes.h"

//Cuda checks
__global__ void matrix_multiply_kernel(unsigned char *temp, unsigned char *matrix, float *kernal, int order, int middle, int windowSizeX, int windowSizeY){
//Find place in the execution
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;
float sum = 0.0;
//If out of bounds, do nothing
if(y >= windowSizeY || x >= windowSizeX){
return;
}
//Else do function
for(int y2 = 0; y2 < order; y2++){
for(int x2 = 0; x2 < order; x2++){
int tempX = x - middle + x2, tempY = y - middle + y2;
if(tempX < 0){
tempX = 0;
}else if(tempX >= windowSizeX){
tempX = windowSizeX - 1;
}
if(tempY < 0){
tempY = 0;
}else if(tempY >= windowSizeY){
tempY = windowSizeY - 1;
}
sum += temp[(windowSizeX * tempY) + tempX] * kernal[(order * x2) + y2];
}
}
//Clamp the sum value
if(sum < 0){
sum = 0;
}else if(sum > 255){
sum = 255;
}
//Add sum value to matrix
matrix[(windowSizeX * y) + x] = (unsigned char) sum;

}