#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_gray(unsigned char *input, int offset, int streamSize, unsigned char* gray, int size) {

int gray_idx = (offset/3) + (blockIdx.x * blockDim.x + threadIdx.x);
int rgb_idx = (offset) + ((blockIdx.x * blockDim.x + threadIdx.x) * 3);

if (((blockIdx.x * blockDim.x + threadIdx.x)*3)>=streamSize || gray_idx>=size) {
return;
}

gray[gray_idx] = (gray_value[0] * input[rgb_idx]) + (gray_value[1] * input[rgb_idx + 1]) + (gray_value[2] * input[rgb_idx + 2]);
}