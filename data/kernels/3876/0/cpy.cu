#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void cpy(float *a, float *b, int n) {
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n)
a[i] = b[i];
}