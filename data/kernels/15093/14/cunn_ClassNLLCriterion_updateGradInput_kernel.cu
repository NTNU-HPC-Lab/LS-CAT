#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget, float* weights, bool apply_weights) {
register int i, j, t;
for (i = threadIdx.x; i < nframe; i += NTHREADS) {
for (j = 0; j < ntarget; ++j) {
t = (int)target[i * ntarget + j] - 1;
if (t >= 0) {
if (apply_weights) {
gradInput[i * ndim + t] = grad * weights[t];
} else {
gradInput[i * ndim + t] = grad;
}
}
}
}
}