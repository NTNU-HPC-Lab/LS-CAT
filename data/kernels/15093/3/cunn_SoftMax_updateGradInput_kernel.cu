#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
__shared__ float buffer[SOFTMAX_THREADS];
int k = blockIdx.x;
float *gradInput_k = gradInput + k*dim;
float *output_k = output + k*dim;
float *gradOutput_k = gradOutput + k*dim;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

// sum?
buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
buffer[threadIdx.x] += gradOutput_k[i] * output_k[i];

__syncthreads();

// reduce
if (threadIdx.x == 0)
{
float sum_k = 0;
for (int i=0; i<blockDim.x; i++)
sum_k += buffer[i];
buffer[0] = sum_k;
}

__syncthreads();

float sum_k = buffer[0];
for (int i=i_start; i<i_end; i+=i_step)
gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
}