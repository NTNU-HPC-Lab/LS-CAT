#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateError(float *aFourth, float *err, int expectedOutput)
{
int i = threadIdx.x;
err[i] = aFourth[i] - (i + 1 == expectedOutput);
}