#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void ReductionMin(unsigned int *sdata, unsigned int *results, int n)    //take thread divergence into account
{
// extern __shared__ int sdata[];
unsigned int tx = threadIdx.x;

// block-wide reduction
for(unsigned int offset = blockDim.x>>1; offset > 0; offset >>= 1)
{
__syncthreads();
if(tx < offset)
{
if(sdata[tx + offset] < sdata[tx] || sdata[tx] == 0)
sdata[tx] = sdata[tx + offset];
}

}

// finally, thread 0 writes the result
if(threadIdx.x == 0)
{
// the result is per-block
*results = sdata[0];
}
}