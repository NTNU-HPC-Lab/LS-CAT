#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid_derivate(float x){
return __fmul_rn(x, __fsub_rn(1.0f, x));
}
__device__ float sigmoid(float x){
return __frcp_rn(__fadd_rn(1, exp(-x)));
}
__global__ void sigmoidBackward(float* R, float* V, int x, int y){
int index = blockDim.x * blockIdx.x + threadIdx.x;
if(index < x*y)
R[index] = sigmoid_derivate(V[index]);
}