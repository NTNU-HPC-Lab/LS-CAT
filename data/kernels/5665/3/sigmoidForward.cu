#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float x){
return __frcp_rn(__fadd_rn(1, exp(-x)));
}
__global__ void sigmoidForward(float* R, float* V, int x, int y){
int index = blockDim.x * blockIdx.x + threadIdx.x;
if(index < x*y)
R[index] = sigmoid(V[index]);
}