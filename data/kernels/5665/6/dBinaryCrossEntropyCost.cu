#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int x) {
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < x) {
dY[index] = -1.0 * ( target[index]/predictions[index] - ((1 - target[index])/(1 - predictions[index])) );
}
}