#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binaryCrossEntropyCost(float* cost, float* predictions, float* target, int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size) {
float partial_cost = target[index] * logf(1.0e-15+predictions[index])
+ (1.0f - target[index]) * logf(1.0e-15+(1.0f - predictions[index]));
atomicAdd(cost, -1.0 * partial_cost / size);
}
}