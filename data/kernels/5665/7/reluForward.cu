#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluForward(float* R, float* V, int x, int y){
int index = blockDim.x * blockIdx.x + threadIdx.x;
if(index < x*y)
R[index] = fmaxf(V[index], 0);
}