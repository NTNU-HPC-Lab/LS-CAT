#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluBackward(float* dZ, float* top_diff, float* V, int x, int y){
int index = blockDim.x * blockIdx.x + threadIdx.x;
if(index < x*y){
if(V[index] > 0) {
dZ[index] = top_diff[index];
}else{
dZ[index] = 0;
}
}
}