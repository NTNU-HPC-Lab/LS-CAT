#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif



#ifdef __cplusplus
}
#endif
__global__ void game(int* A, const int N, const int largeur, const int hauteur){
int idx = blockDim.x * blockIdx.x + threadIdx.x;

int y = idx / hauteur;
int x = idx - (y * largeur);
if (y >= hauteur || x >= largeur)
return;

int me = A[idx];
int north =  0 ;
int northEast = 0;
int northWest = 0;
int south = 0;
int southEast = 0;
int southWest = 0;
int east = 0;
int west = 0;
if (x > 0)
west = A[idx -1];
if (x < largeur - 1)
east = A[idx + 1];
if (y > 0)
north = A[idx - largeur];
if (y < hauteur - 1)
south = A[idx + largeur];

if ((y < hauteur - 1) && (x < largeur - 1))
southEast =  A[idx + largeur + 1];
if ((y < hauteur - 1) && (x > 0))
southWest =  A[idx + largeur - 1];
if ((y > 0) && (x >0))
northWest =  A[idx - largeur - 1];
if ((y > 0) && (x < largeur - 1))
northEast =  A[idx - largeur + 1];
int res = north + south + east + west + northEast + northWest + southEast + southWest;
//__syncthreads();
if ((me == 1) && (res < 2) || (res > 3))
A[idx] = 0;
else
if ((me == 0) &&  (res == 3))
A[idx] = 1;
}