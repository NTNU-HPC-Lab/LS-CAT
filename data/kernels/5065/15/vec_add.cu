#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif






#ifdef __cplusplus
}
#endif
__global__ void vec_add(float *A, float *B, float* C, int size)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;

if(index<size)
C[index] = A[index] + B[index];
}