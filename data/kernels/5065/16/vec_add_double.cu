#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif






#ifdef __cplusplus
}
#endif
__global__ void vec_add_double(double *A, double *B, double* C, int size)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;

if(index<size)
C[index] = A[index] + B[index];
}