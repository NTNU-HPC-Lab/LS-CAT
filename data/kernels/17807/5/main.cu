#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "frontier_init_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *p_frontier_tail_d = NULL;
hipMalloc(&p_frontier_tail_d, XSIZE*YSIZE);
int *c_frontier_tail_d = NULL;
hipMalloc(&c_frontier_tail_d, XSIZE*YSIZE);
int *p_frontier_d = NULL;
hipMalloc(&p_frontier_d, XSIZE*YSIZE);
int *visited_d = NULL;
hipMalloc(&visited_d, XSIZE*YSIZE);
int *label_d = NULL;
hipMalloc(&label_d, XSIZE*YSIZE);
int source = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
frontier_init_kernel<<<gridBlock,threadBlock>>>(p_frontier_tail_d,c_frontier_tail_d,p_frontier_d,visited_d,label_d,source);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
frontier_init_kernel<<<gridBlock,threadBlock>>>(p_frontier_tail_d,c_frontier_tail_d,p_frontier_d,visited_d,label_d,source);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
frontier_init_kernel<<<gridBlock,threadBlock>>>(p_frontier_tail_d,c_frontier_tail_d,p_frontier_d,visited_d,label_d,source);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}