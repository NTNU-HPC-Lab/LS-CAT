#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void frontier_init_kernel(int* p_frontier_tail_d, int* c_frontier_tail_d, int* p_frontier_d, int* visited_d, int* label_d, int source) {
visited_d[source] = 1;
*c_frontier_tail_d = 0;
p_frontier_d[0] = source;
*p_frontier_tail_d = 1;
label_d[source] = 0;
}