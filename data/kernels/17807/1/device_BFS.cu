#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_BFS(const int* edges, const int* dests, int* labels, int* visited, int* c_frontier_tail, int* c_frontier, int* p_frontier_tail, int* p_frontier) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < *p_frontier_tail) {
int c_vertex = p_frontier[index];
for (int i = edges[c_vertex]; i < edges[c_vertex+1]; i++) {
int was_visited = atomicExch(visited + dests[i], 1);
if (!was_visited) {
int old_tail = atomicAdd(c_frontier_tail, 1);
c_frontier[old_tail] = dests[i];
labels[dests[i]] = labels[c_vertex] + 1;
}
}
}
}