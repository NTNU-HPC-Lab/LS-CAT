#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void BFS_UNIFIED(int source, int* edges, int* dest, int* label, int* visited, int *c_frontier_tail, int *c_frontier, int *p_frontier_tail, int *p_frontier)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < *p_frontier_tail)
{
int c_vertex = p_frontier[i];
for (int i = edges[c_vertex]; i < edges[c_vertex+1]; ++i)
{
int was_visited = atomicExch(visited + dest[i], 1);
if (!was_visited)
{
int old_tail = atomicAdd(c_frontier_tail, 1);
c_frontier[old_tail] = dest[i];
label[dest[i]] = label[c_vertex] + 1;
}
}

}
}