#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_Bqueue_kernel(unsigned int* p_frontier, unsigned int* p_frontier_tail, unsigned int* c_frontier, unsigned int* c_frontier_tail, unsigned int* edges, unsigned int* dest, unsigned int* label, unsigned int* visited) {
__shared__ unsigned int c_frontier_s[BLOCK_QUEUE_SIZE];
__shared__ unsigned int c_frontier_tail_s;
__shared__ unsigned int our_c_frontier_tail;

if (threadIdx.x == 0) {
c_frontier_tail_s = 0;
}
__syncthreads();

const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < *p_frontier_tail) {
const unsigned int my_vertex = p_frontier[tid];
for (unsigned int i = edges[my_vertex]; i < edges[my_vertex + 1]; ++i) {
const unsigned int was_visited = atomicExch(&(visited[dest[i]]), 1);
if (not was_visited) {
label[dest[i]] = label[my_vertex];
const unsigned int my_tail = atomicAdd(&c_frontier_tail_s, 1);
if (my_tail < BLOCK_QUEUE_SIZE) {
c_frontier_s[my_tail] = dest[i];
} else {
c_frontier_tail_s = BLOCK_QUEUE_SIZE;
const unsigned int my_global_tail = atomicAdd(c_frontier_tail, 1);
c_frontier[my_global_tail] = dest[i];
}
}
}
__syncthreads();

if (threadIdx.x == 0) {
our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
}
__syncthreads();

for (unsigned int i = threadIdx.x; i < c_frontier_tail_s; i+= blockDim.x) {
c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
}
}
}