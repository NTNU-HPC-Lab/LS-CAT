#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Correlation_forward( float *output, int nOutputChannels, int outputHeight, int outputWidth, float *rInput1, int nInputChannels, int inputHeight, int inputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
// n (batch size), c (num of channels), y (height), x (width)

int pInputWidth = inputWidth + 2 * pad_size;
int pInputHeight = inputHeight + 2 * pad_size;

int kernel_rad = (kernel_size - 1) / 2;
int displacement_rad = max_displacement / stride2;
int displacement_size = 2 * displacement_rad + 1;

int n  = blockIdx.x;
int y1 = blockIdx.y * stride1 + max_displacement + kernel_rad;
int x1 = blockIdx.z * stride1 + max_displacement + kernel_rad;
int c = threadIdx.x;

int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
int pdimxc = pInputWidth * nInputChannels;
int pdimc = nInputChannels;

int tdimcyx = nOutputChannels * outputHeight * outputWidth;
int tdimyx = outputHeight * outputWidth;
int tdimx = outputWidth;

float nelems = kernel_size * kernel_size * pdimc;

__shared__ float prod_sum[THREADS_PER_BLOCK];

// no significant speed-up in using chip memory for input1 sub-data,
// not enough chip memory size to accomodate memory per block for input2 sub-data
// instead i've used device memory for both

// element-wise product along channel axis
for (int tj = -displacement_rad; tj <= displacement_rad; ++tj ) {
for (int ti = -displacement_rad; ti <= displacement_rad; ++ti ) {
prod_sum[c] = 0;
int x2 = x1 + ti*stride2;
int y2 = y1 + tj*stride2;

for (int j = -kernel_rad; j <= kernel_rad; ++j) {
for (int i = -kernel_rad; i <= kernel_rad; ++i) {
for (int ch = c; ch < pdimc; ch += THREADS_PER_BLOCK) {
int indx1 = n * pdimyxc + (y1+j) * pdimxc + (x1 + i) * pdimc + ch;
int indx2 = n * pdimyxc + (y2+j) * pdimxc + (x2 + i) * pdimc + ch;

prod_sum[c] += rInput1[indx1] * rInput2[indx2];
}
}
}

// accumulate
__syncthreads();
if (c == 0) {
float reduce_sum = 0;
for (int index = 0; index < THREADS_PER_BLOCK; ++index) {
reduce_sum += prod_sum[index];
}
int tc = (tj + displacement_rad) * displacement_size + (ti + displacement_rad);
const int tindx = n * tdimcyx + tc * tdimyx + blockIdx.y * tdimx + blockIdx.z;
output[tindx] = reduce_sum / nelems;
}

}
}

}