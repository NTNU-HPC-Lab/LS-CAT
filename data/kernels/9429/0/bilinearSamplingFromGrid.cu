#include "hip/hip_runtime.h"
#include "includes.h"
// Bilinear sampling is done in BHWD (coalescing is not obvious in BDHW)
// we assume BHWD format in inputImages
// we assume BHW(YX) format on grids

__device__ bool between(int value, int lowerBound, int upperBound)
{
return (value >= lowerBound && value <= upperBound);
}
__device__ void getTopLeft(float x, int width, int& point, float& weight)
{
/* for interpolation :
stores in point and weight :
- the x-coordinate of the pixel on the left (or y-coordinate of the upper pixel)
- the weight for interpolating
*/

float xcoord = (x + 1) * (width - 1) / 2;
point = floor(xcoord);
weight = 1 - (xcoord - point);
}
__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth, float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth, float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight, int output_strideWidth, int inputImages_channels, int inputImages_height, int inputImages_width, int output_width)
{
// each (32,16) block 16 output pixels (for coalescing the grid read)
// x,y = coordinates (xOut = blockIdx.x*16+blockDim.y+threadIdx.y)
// z = batch index
// threadIdx.x : used for features (coalescing is trivial)

const int xOut = blockIdx.x*blockDim.y+threadIdx.y;
const bool withinImageBounds = xOut < output_width;
const bool withinGridBounds = blockIdx.x*blockDim.y + threadIdx.x / 2 < output_width;
const int yOut = blockIdx.y;
const int width = inputImages_width;
const int height = inputImages_height;

const int b = blockIdx.z;

float yf,xf;

__shared__ float gridData[32];
if (threadIdx.y==0 && withinGridBounds)
{
gridData[threadIdx.x] = grids_data[b*grids_strideBatch + yOut*grids_strideHeight + xOut*grids_strideWidth + threadIdx.x];
}
__syncthreads();
if(!withinImageBounds) return;
yf = gridData[threadIdx.y*2];
xf = gridData[threadIdx.y*2+1];

int yInTopLeft, xInTopLeft;
float yWeightTopLeft, xWeightTopLeft;
getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;
const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

float v=0;
float inTopLeft=0;
float inTopRight=0;
float inBottomLeft=0;
float inBottomRight=0;

bool topLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft, 0, height-1);
bool topRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft, 0, height-1);
bool bottomLeftIsIn = between(xInTopLeft, 0, width-1) && between(yInTopLeft+1, 0, height-1);
bool bottomRightIsIn = between(xInTopLeft+1, 0, width-1) && between(yInTopLeft+1, 0, height-1);

// interpolation happens here
for(int t=threadIdx.x; t<inputImages_channels; t+= blockDim.x)
{
if(topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
if(topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
if(bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
if(bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];

v = xWeightTopLeft * yWeightTopLeft * inTopLeft
+ (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
+ xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
+ (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

output_data[outAddress + t] = v;
}
}