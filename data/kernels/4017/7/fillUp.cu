#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Various matrix utils using cuda
**/


/**
* Kronecker product of two matrices kernel
* input :
* a : first matrix
* nax, nay : matrix a dimensions
* b: second matrix
* nbx, nby : matrix b dimensions
* results : kronecker product of a and b
**/

__global__ void fillUp(size_t * d_dst, size_t N){
int myId = blockIdx.x * blockDim.x + threadIdx.x;
if (myId >= N)
return;
d_dst[myId] = myId;
}