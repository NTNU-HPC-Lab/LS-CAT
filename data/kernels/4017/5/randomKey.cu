#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Various matrix utils using cuda
**/


/**
* Kronecker product of two matrices kernel
* input :
* a : first matrix
* nax, nay : matrix a dimensions
* b: second matrix
* nbx, nby : matrix b dimensions
* results : kronecker product of a and b
**/

__global__ void randomKey(size_t N, float * d_dst, unsigned long seed){
int myId = blockIdx.x * blockDim.x + threadIdx.x;
if (myId >= N)
return;
hiprandState state;
hiprand_init ( seed, myId, 0, &state);
float RANDOM = hiprand_uniform( &state );
d_dst[myId] = (float)RANDOM;
}