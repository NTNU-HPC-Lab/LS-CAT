#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Various matrix utils using cuda
**/


/**
* Kronecker product of two matrices kernel
* input :
* a : first matrix
* nax, nay : matrix a dimensions
* b: second matrix
* nbx, nby : matrix b dimensions
* results : kronecker product of a and b
**/

__global__ void reduceSum(double * d_arr, const size_t sz, double * d_out)
{
extern __shared__ double sh_out [];
int myId = threadIdx.x + blockDim.x * blockIdx.x;
int tId = threadIdx.x;
if ( myId >= sz)
{
sh_out[tId] = 0.0;
}
else
{
// Fill in the shared memory
sh_out[tId] = d_arr[myId];
}
__syncthreads();
for  (unsigned int s = blockDim.x /2; s > 0; s >>=1)
{
if (tId < s)
{
sh_out[tId] += sh_out[tId+s];
}
__syncthreads();
}
if (tId == 0)
{
d_out[blockIdx.x] = sh_out[0];
}
}