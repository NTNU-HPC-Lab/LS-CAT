#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findID(double *a, int n){

// First we need to find our global threadID
int tPosX = blockIdx.x * blockDim.x + threadIdx.x;
// Make sure we are not out of range
if (tPosX < n){
a[tPosX] = tPosX;
}
}