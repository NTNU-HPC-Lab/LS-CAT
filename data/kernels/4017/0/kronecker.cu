#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Various matrix utils using cuda
**/


/**
* Kronecker product of two matrices kernel
* input :
* a : first matrix
* nax, nay : matrix a dimensions
* b: second matrix
* nbx, nby : matrix b dimensions
* results : kronecker product of a and b
**/

__global__ void kronecker(double * a, int nax, int nay, double * b, int nbx, int nby, double * result){

// First we need to find our global threadID
int tPosX = blockIdx.x * blockDim.x + threadIdx.x;
int tPosY = blockIdx.y * blockDim.y + threadIdx.y;
int resSzx = nax * nbx;
//int resSzy = nay * nby;
int idxA = floor((tPosX) / (double)nbx);
int idyA = floor((tPosY) / (double)nby);
int idxB = (tPosX) % nbx;
int idyB = (tPosY) % nby;
// Check if the indices are within range
if (idxA >= nax || idyA > nay || idxB > nbx || idyB > nby)
{
result[tPosX + tPosY * resSzx] = -1;
return;
}
// Multiply appropriate elements
result[tPosX + tPosY * resSzx] = a[idyA * nax +  idxA] * b[idyB * nbx + idxB];
}