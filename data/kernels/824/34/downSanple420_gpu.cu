#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void downSanple420_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, uint8_t *downCh1, uint8_t *downCh2, size_t width, size_t height)
{
int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
if (threadCoord.x < width && (threadCoord.y << 1) < height)
{
int2 pixelCoord;
hipTextureObject_t *ch;
uint8_t *downCh;

// Remember thread divergence happens at the wrap level only, that will parallelize well
if (threadCoord.x < (width >> 1))
{
pixelCoord = make_int2(threadCoord.x << 1, threadCoord.y << 1);
ch = &ch1;
downCh = downCh1;
}
else
{
pixelCoord = make_int2((threadCoord.x - (width >> 1)) << 1, threadCoord.y << 1);
ch = &ch2;
downCh = downCh2;
}

int16_t bias = (pixelCoord.x & 1) + 1;
uint16_t pixel = (tex2D<uint16_t>(*ch, pixelCoord.x, pixelCoord.y) + tex2D<uint16_t>(*ch, pixelCoord.x + 1, pixelCoord.y)
+ tex2D<uint16_t>(*ch, pixelCoord.x, pixelCoord.y + 1) + tex2D<uint16_t>(*ch, pixelCoord.x + 1, pixelCoord.y + 1) + bias) >> 2;
downCh[((pixelCoord.y >> 1) * width + pixelCoord.x) >> 1] = (uint8_t)pixel;
}
}