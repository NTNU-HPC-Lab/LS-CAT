#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void downSanple422_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, uint8_t *downCh1, uint8_t *downCh2, size_t width, size_t height)
{
int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
if (threadCoord.x < width && threadCoord.y < height)
{
int2 pixelCoord;
hipTextureObject_t *ch;
uint8_t *downCh;

// Remember thread divergence happens at the wrap level only, that will parallelize well
if (threadCoord.x < (width >> 1))
{
pixelCoord = make_int2(threadCoord.x << 1, threadCoord.y);
ch = &ch1;
downCh = downCh1;
}
else
{
pixelCoord = make_int2((threadCoord.x - (width >> 1)) << 1, threadCoord.y);
ch = &ch2;
downCh = downCh2;
}

int16_t bias = pixelCoord.x & 1;
uint16_t pixel = (tex2D<uint16_t>(*ch, pixelCoord.x, pixelCoord.y) + tex2D<uint16_t>(*ch, pixelCoord.x + 1, pixelCoord.y) + bias) >> 1;
downCh[(pixelCoord.y * width + pixelCoord.x) >> 1] = (uint8_t)pixel;
}
}