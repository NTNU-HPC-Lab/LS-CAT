#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shiftLeftPixels(int16_t *bayImg, size_t width, size_t height, int bppMult)
{
int2 pixelCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

if (pixelCoord.x < width && pixelCoord.y < height)
{
bayImg[pixelCoord.y * width + pixelCoord.x] <<= bppMult;
}
}