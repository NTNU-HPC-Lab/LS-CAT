#include "hip/hip_runtime.h"
#include "includes.h"

#define KERNEL_RADIUS 31
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[ KERNEL_LENGTH ];

__global__ void convolutionX_63_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int outofbounds, float outofboundsvalue )
{
__shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z;

const int firstPixelInLine = ROWS_BLOCKDIM_X * ROWS_HALO_STEPS - threadIdx.x;
const int lastPixelInLine = imageW - baseX - 1;

// set the input and output arrays to the right offset (actually the output is not at the right offset, but this is corrected later)
d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

// Load main data
// Start copying after the ROWS_HALO_STEPS, only the original data that will be convolved
#pragma unroll

for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : outofboundsvalue;
else
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[ lastPixelInLine ];
}

// Load left halo
// If the data fetched is outside of the image (note: baseX can be <0 for the first block) , use a zero-out of bounds strategy
#pragma unroll

for (int i = 0; i < ROWS_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : outofboundsvalue;
else
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[ firstPixelInLine ];
}

//Load right halo
#pragma unroll

for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : outofboundsvalue;
else
s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[ lastPixelInLine ];
}

//Compute and store results
__syncthreads();

// this pixel is not part of the image and does not need to be convolved
if ( baseY >= imageH )
return;

#pragma unroll

for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
{
if (imageW - baseX > i * ROWS_BLOCKDIM_X)
{
float sum = 0;

#pragma unroll

for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
{
sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
}

d_Dst[i * ROWS_BLOCKDIM_X] = sum;
}
}
}