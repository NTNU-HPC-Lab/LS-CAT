#include "hip/hip_runtime.h"
#include "includes.h"

#define KERNEL_RADIUS 31
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[ KERNEL_LENGTH ];

__global__ void convolutionZ_63_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int outofbounds, float outofboundsvalue )
{
// here it is [x][z], we leave out y as it has a size of 1
__shared__ float s_Data[DEPTH_BLOCKDIM_X][(DEPTH_RESULT_STEPS + 2 * DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + 1];

//Offset to the upper halo edge
const int baseX = blockIdx.x * DEPTH_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y;
const int baseZ = (blockIdx.z * DEPTH_RESULT_STEPS - DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + threadIdx.z;

const int firstPixelInLine = (DEPTH_BLOCKDIM_Z * DEPTH_HALO_STEPS - threadIdx.z) * imageW * imageH;
const int lastPixelInLine = (imageD - baseZ - 1) * imageW * imageH;

d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

//Main data
#pragma unroll

for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : d_Src[ lastPixelInLine ];
}

//Upper halo
#pragma unroll

for (int i = 0; i < DEPTH_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (baseZ >= -i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (baseZ >= -i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (baseZ >= -i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : d_Src[ firstPixelInLine ];
}

//Lower halo
#pragma unroll

for (int i = DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS + DEPTH_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z]= (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z]= (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z]= (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : d_Src[ lastPixelInLine ];
}

//Compute and store results
__syncthreads();

// this pixel is not part of the image and does not need to be convolved
if ( baseX >= imageW )
return;

#pragma unroll

for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
{
if (imageD - baseZ > i * DEPTH_BLOCKDIM_Z)
{
float sum = 0;

#pragma unroll

for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
{
sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z + j];
}

d_Dst[i * DEPTH_BLOCKDIM_Z * imageW * imageH] = sum;
}
}
}