#include "hip/hip_runtime.h"
#include "includes.h"

#define KERNEL_RADIUS 31
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[ KERNEL_LENGTH ];

__global__ void convolutionY_63_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int outofbounds, float outofboundsvalue )
{
__shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

//Offset to the upper halo edge
const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
const int baseZ = blockIdx.z;

const int firstPixelInLine = (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS - threadIdx.y) * imageW;
const int lastPixelInLine = (imageH - baseY - 1) * imageW;

d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

//Main data
#pragma unroll

for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : d_Src[ lastPixelInLine ];
}

//Upper halo
#pragma unroll

for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : d_Src[ firstPixelInLine ];
}

//Lower halo
#pragma unroll

for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
{
if ( outofbounds == 0 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : 0;
else if ( outofbounds == 1 )
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : outofboundsvalue;
else
s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : d_Src[ lastPixelInLine ];
}

//Compute and store results
__syncthreads();

// this pixel is not part of the image and does not need to be convolved
if ( baseX >= imageW )
return;

#pragma unroll

for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
{
if (imageH - baseY > i * COLUMNS_BLOCKDIM_Y)
{
float sum = 0;

#pragma unroll

for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
{
sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
}

d_Dst[i * COLUMNS_BLOCKDIM_Y * imageW] = sum;
}
}
}