#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MergeRank(float * d_input, float * d_output)
{
int indexA = blockIdx.x * blockDim.x + threadIdx.x;
int indexB = indexA + 2048;
float temp1 = d_input[indexA];
float temp2 = d_input[indexB];
int indexAB = 2048;
while (d_input[indexAB] < temp1) {
indexAB++;
}
int indexBA = 0;
while (d_input[indexBA] < temp2) {
indexBA++;
}
__syncthreads();
d_output[indexA + indexAB + 1] = temp1;
d_output[indexB + indexBA + 1] = temp2;

}