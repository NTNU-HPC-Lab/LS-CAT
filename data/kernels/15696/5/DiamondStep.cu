#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DiamondStep(uint8_t* matrix, unsigned *random, int currentSize, int matrixSize, int randValue)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int half = currentSize / 2;
int minRand = -randValue;
int row = y * currentSize + half;
int col = x * currentSize + half;
int value;
value = (matrix[(row - half)*matrixSize + (col - half)] +
matrix[(row - half)*matrixSize + (col + half)] +
matrix[(row + half)*matrixSize + (col - half)] +
matrix[(row + half)*matrixSize + (col + half)] +
//VERSION 1
//(random[x*gridDim.x+y] % (randValue - minRand) + minRand)) / 4;
//VERSION 2
(random[row*matrixSize + col] % (randValue - minRand) + minRand)) / 4;
matrix[row*matrixSize + col] = value;
}