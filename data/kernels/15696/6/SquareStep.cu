#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SquareStep(uint8_t* matrix, unsigned* random, int currentSize, int matrixSize, int maxRowThread, int maxColThread, int randValue)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
/*x/y can be greater than maxRowThread/maxColThread because the number
of created threads might not be multiple of the number of threads in a block*/
if (x < maxRowThread && y < maxColThread)
{
int half = currentSize / 2;
int minRand = -randValue;
int value = 0;
int div = 0;
int cond;
int elemX = x * currentSize*(y % 2 == 0) +
y * half*(y % 2 != 0);
int elemY = (y*half + half)*(y % 2 == 0) +
x * currentSize*(y % 2 != 0);
// CUDA VERSION 2: it uses conditions as variables
// to avoid divergent branches
cond = elemX != 0;
value += matrix[(elemX - half * cond) *
matrixSize + elemY] * cond;
div += cond;
cond = elemX != matrixSize - 1;
value += matrix[(elemX + half * cond) *
matrixSize + elemY] * cond;
div += cond;
cond = elemY != 0;
value += matrix[elemX * matrixSize +
elemY - half * cond] * cond;
div += cond;
cond = elemY != matrixSize - 1;
value += matrix[elemX*matrixSize + elemY
+ half * cond] * cond;
div += cond;
/*
// CUDA VERSION 1: it uses divergent branches
if (elemX != 0)
{
value += matrix[(elemX - half)*matrixSize + elemY];
div++;
}
if (elemX != matrixSize-1)
{
value += matrix[(elemX + half)*matrixSize + elemY];
div++;
}
if (elemY != 0)
{
value += matrix[elemX*matrixSize + elemY - half];
div++;
}
if (elemY != matrixSize-1)
{
value += matrix[elemX*matrixSize + elemY + half];
div++;
}*/
//VERSION 1: random index is correct for the
//first version of random generation but not for the second one
//value += (minRand + random[x*gridDim.x+y] % (randValue - minRand));
//VERSION 2
value += (minRand + random[elemX*matrixSize+elemY] % (randValue - minRand));
matrix[elemX*matrixSize + elemY] = value / div;
}
}