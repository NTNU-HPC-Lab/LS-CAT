#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_MonoBlock( unsigned char *buffer,long size,unsigned int *histo ) {

__shared__ unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x , offset = blockDim.x;
while (i < size) {
atomicAdd( &temp[buffer[i]], 1);
i += offset;
}
__syncthreads();

atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );

}