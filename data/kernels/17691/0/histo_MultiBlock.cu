#include "hip/hip_runtime.h"
#include "includes.h"
#define SIZE 2048*1024
#define BLOCKS 1000
#define THREADS 256

__global__ void histo_MultiBlock( unsigned char *buffer,long size,unsigned int *histo ) {

__shared__ unsigned int temp[256];
int i = threadIdx.x + blockIdx.x * THREADS;
int offset= THREADS * BLOCKS;
int memoffset = blockIdx.x * THREADS;
if(threadIdx.x <256)
temp[threadIdx.x] = 0;
__syncthreads();

while(i<size){
atomicAdd( &temp[buffer[i]], 1);
i+=offset;
}
__syncthreads();
if(threadIdx.x <256)
atomicAdd( &(histo[threadIdx.x+memoffset]), temp[threadIdx.x] );
}