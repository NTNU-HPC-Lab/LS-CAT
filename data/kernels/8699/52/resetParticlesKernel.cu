#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resetParticlesKernel(float3* pos, float3* vel, float* age, float* life, char* opafix, int nParts)
{
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

int n = x;

if (n<nParts) {
pos[n] = make_float3(2.0,0.0,0.0);
vel[n] =  make_float3(0.0,0.0,0.0);
age[n] = -1.0;
life[n] = -1.0;
opafix[n] = 0;
}
}