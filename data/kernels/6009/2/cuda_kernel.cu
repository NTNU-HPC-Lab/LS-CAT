#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_kernel(double *A, double *B, double *C, int arraySize) {
// Get thread ID.
int tid = blockDim.x * blockIdx.x + threadIdx.x;

// Check if thread is within array bounds.
if (tid < arraySize) {
// Add a and b.
C[tid] = A[tid] + B[
tid];
}
}