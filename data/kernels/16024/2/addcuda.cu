#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f




__global__ void addcuda(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
for (int j = 0; j < 10; j++) {
int x = blockIdx.x;
int y = threadIdx.x;
int i = x * K * D + y * D;

float Px = P[i + 0];
float Py = P[i + 1];
float Pz = P[i + 2];
float E = Eg[i/3];
float Ep = Epg[i/3];

float Qx = Q[i + 0];
float Qy = Q[i + 1];
float Qz = Q[i + 2];

float nQx = Q[i + 0] + c * P[i + 0];
float nQy = Q[i + 1] + c * P[i + 1];
float nQz = Q[i + 2] + c * P[i + 2];

// Îòðàæåíèå îò ñòåíîê îáëàñòè

if ((nQx > 1) || (nQx < 0)) {
Px = (-1) * Px;
}
if ((nQy > 1) || (nQy < 0)) {
Py = (-1) * Py;
}
if ((nQz > 1) || (nQz < 0)) {
Pz = (-1) * Pz;
}

// Îòðàæåíèå îò òóðáóëåíòíîñòåé

for (int nt = 0; nt < Nt; nt += 1) {
float Range = (sqrt(pow(Qx - Qt[nt + 0], 2) + pow(Qy - Qt[nt + 1], 2) + pow(Qz - Qt[nt + 2], 2)));
float nRange = (sqrt(pow(nQx - Qt[nt + 0], 2) + pow(nQy - Qt[nt + 1], 2) + pow(nQz - Qt[nt + 2], 2)));

if((Range > Rt) && (nRange < Rt)) {
float DirX = (nQx - Qt[nt + 0]) / Range;
float DirY = (nQy - Qt[nt + 1]) / Range;
float DirZ = (nQz - Qt[nt + 2]) / Range;
float PnormKoe = ((Px * DirX) + (Py * DirY) + (Pz * DirZ));
float Pnormt = ((Pt[nt + 0] * DirX) + (Pt[nt + 1] * DirY) + (Pt[nt + 2] * DirZ));
E -= (ct / c) * (PnormKoe * PnormKoe) * (Pnormt * abs(Pnormt));
Px -= 2 * DirX;
Py -= 2 * DirY;
Pz -= 2 * DirZ;
}
}
// ×àñòèöà âûëåòàåò èç îáëàñòè, çàïèñûâàåòñÿ åå ýíåðãèÿ è ñáðàñûâàåòñÿ äî íà÷àëüíîãî çíà÷åíèÿ.
// ×àñòèöà ïðîäîëæàåò äâèãàòüñÿ ïî òðàåêòîðèè
// Ep ñëó÷àéíàÿ âåëè÷èíà ëèíåéíî çàâèñÿùàÿ îò ýíåðãèè
if ((nQz > 1) && (E > Ep)) {
E = 100.0f;
}
// Àäèàáàòè÷åñêîå îõëàæäåíèå
if (nQz > 0.5) {
E -= 0.0001f;
}
//Ïðèðàùåíèå ýíåðãèè ïðè ïåðåñå÷åíèè öåíòðà
if (((nQz > 0.5f) && (Qz < 0.5f)) || ((Qz > 0.5f) && (nQz < 0.5f))) {
E += 1.0f;
}

// Çàïèñü â ïàìÿòü
Q[i + 0] = nQx;
Q[i + 1] = nQy;
Q[i + 2] = nQz;

P[i + 0] = Px;
P[i + 1] = Py;
P[i + 2] = Pz;
Eg[i/3] = E;
}
}