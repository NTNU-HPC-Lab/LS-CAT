#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f




__global__ void NextQTur(float* Qt, float* Pt) {
int i = threadIdx.x;
Qt[i + 0] += Pt[i + 0] * ct;
Qt[i + 1] += Pt[i + 1] * ct;
Qt[i + 2] += Pt[i + 2] * ct;
}