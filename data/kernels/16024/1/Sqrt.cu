#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f




__global__ void Sqrt(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
int x = blockIdx.x;
int y = threadIdx.x;
int i = x * K * D + y * D;
//int z = threadIdx.z;
//printf("I = %i \n", x);
for (int j = 0; j < 3; j++) {
Q[i + j] = 0.01;
Qt[i + j] = 0.6;
P[i + j] = 0.3;
Pt[i + j] = 0.5;
Epg[i / D ] = 100000;
Eg[i / D ] = 0.5;
}
}