#include "hip/hip_runtime.h"
#include "includes.h"



#define NUMAR_NODURI 500

#define NUMAR_MUCHII 500

#define COST_MAXIM 1000000

typedef struct
{
int nod1;
int nod2;
} Muchie;

typedef struct
{
int nodId;
bool vizitat;
} Nod;

//Gaseste costul drumului de la nodul start la nodul stop
__global__ void UpdateCostDrumuri(Nod *noduri, int *costuriTemporale, int *costuriFinale)
{
int nod = threadIdx.x;
if (costuriTemporale[nod] > costuriFinale[nod])
{
costuriTemporale[nod] = costuriFinale[nod];
noduri[nod].vizitat = false;
}
costuriFinale[nod] = costuriTemporale[nod];
}