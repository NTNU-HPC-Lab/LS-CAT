#include "hip/hip_runtime.h"
#include "includes.h"



#define NUMAR_NODURI 500

#define NUMAR_MUCHII 500

#define COST_MAXIM 1000000

typedef struct
{
int nod1;
int nod2;
} Muchie;

typedef struct
{
int nodId;
bool vizitat;
} Nod;

//Gaseste costul drumului de la nodul start la nodul stop
__device__ __host__ int CautareMuchie(Nod start, Nod stop, Muchie *muchii, int *costuri)
{
for (int i = 0; i < NUMAR_MUCHII; i++)
if (muchii[i].nod1 == start.nodId && muchii[i].nod2 == stop.nodId)
return costuri[i];

return COST_MAXIM;
}
__global__ void Cauta_Nod(Nod *noduri, Muchie *muchii, int *costuri, int *costTemporal, int *costFinal)
{
int nod = threadIdx.x;
if (noduri[nod].vizitat == false)
{
noduri[nod].vizitat = true;
for (int n = 0; n < NUMAR_NODURI; n++)
{
//Cauta costul muchiei
int cost = CautareMuchie(noduri[nod], noduri[n], muchii, costuri);

//ia costul minim
if (costFinal[n] > costTemporal[nod] + cost && cost < COST_MAXIM)
costFinal[n] = costTemporal[nod] + cost;
}
}
}