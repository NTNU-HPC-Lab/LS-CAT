#include "hip/hip_runtime.h"
#include "includes.h"


/* Program Parameters */
#define MAXN 15000  /* Max value of N */
#define TILE_WIDTH 32  /* Width of each block */
int N;  /* Matrix size */

/* Matrices */
float overall;
char buffer[10000];
char *pbuff;
int *classIdArray = (int *)malloc(sizeof(int)*26);
int *groupIdArray = (int *)malloc(sizeof(int)*26);
int *roomIdArray =(int *) malloc(sizeof(int)*26);
int *roomSizeAsArray = (int *)malloc(sizeof(int)*5);
int *groupSizeAsArray = (int *)malloc(sizeof(int)*11);
int *timeSlotIdArray = (int *)malloc(sizeof(int)*26);
int *profIdArray = (int *)malloc(sizeof(int)*26);
int *clashes=(int *)calloc(26*26,sizeof(int));

int   *dclassIdArray,*dgroupIdArray,*droomIdArray,*droomSizeAsArray,*dgroupSizeAsArray,*dtimeSlotIdArray,*dprofIdArray,*dclashes;

/* junk */
#define randm() 4|2[uid]&3


/* returns a seed for srand based on the time */
__global__ void calculateFitness(int *classIds, int *roomIds, int *roomCapacities, int *groupIds, int *groupSizes, int *timeSlotIds, int *profIds, int *clashes)
{
int i = threadIdx.x;
int j = blockIdx.x ;

if(classIds[i]==classIds[j] && roomCapacities[classIds[i]]<groupSizes[groupIds[classIds[i]]])
clashes[i*j+i]++;

if (roomIds[classIds[j]] == roomIds[classIds[i]] && timeSlotIds[classIds[i]] == timeSlotIds[classIds[j]]
&& classIds[i] != classIds[j]) {
clashes[i*j+i]++;
}

if (profIds[classIds[j]] == profIds[classIds[j]] && timeSlotIds[classIds[i]] == timeSlotIds[classIds[j]]
&& classIds[i] != classIds[j]) {
clashes[i*j+i]++;
}
}