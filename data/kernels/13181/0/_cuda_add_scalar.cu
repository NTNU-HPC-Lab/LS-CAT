#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32 // # of threads that are executed together (constant valid on most hardware)

/* Simple CUDA example showing:
1) how to sum the values of an array in parallel
2) how to add a scaler to values of an array in parallel
3) how to query GPU hardware

Compile with minimum archetecture specification of 30. Example:
nvcc example.cu - o example -arch=sm_30

Author: Jordan Bonilla
*/

// Allow timing of functions
clock_t start,end;

/* Add "scalar" to every element of the input array in parallel */

// CPU entry point for kernel to add "scalar" to every element of the input array
__global__ void _cuda_add_scalar(int *in, int scalar, int n)
{
int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
while(globalIdx < n)
{
in[globalIdx] = in[globalIdx] + scalar;
globalIdx += blockDim.x * gridDim.x;
}
}