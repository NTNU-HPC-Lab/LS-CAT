#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult(int* results, int* data, int* vec) {
int index = blockIdx.x * blockDim.x  + threadIdx.x;
int result_val = 0;
for(int i = 0; i < cuda_features; i++) {
result_val += vec[i] * data[(index * cuda_features) + i];
}
results[index] = result_val;
}