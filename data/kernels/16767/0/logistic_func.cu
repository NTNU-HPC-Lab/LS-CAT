#include "hip/hip_runtime.h"
#include "includes.h"

//FILE IO RELATED
//max number of lines in the training dataset
#define MAX_ROWS_TRAINING 16896
// max number of columns/features in the training dataset
#define MAX_COLUMNS_TRAINING 26
// max number of rows in the testing dataset
#define MAX_ROWS_TESTING 4096
// max number of columns in the testing data
#define MAX_COLUMNS_TESTING 26
//max number of characters/line
#define MAX_CHAR 300

__constant__ int features = 26;
__constant__ int num_rows = 16896;

long mem_cpy_time = 0;
long beta_cpy_time = 0;

// parallelized across the rows

// parallelized across the features

__global__ void logistic_func(float* log_func_v, float* betas, float* data) {
int row_index = blockIdx.x * blockDim.x + threadIdx.x;
float temp = 0;
for(int j = 0; j < features; j++) {
float accessed_data = data[(row_index * features) + j];
temp += betas[j] * accessed_data;
}
log_func_v[row_index] = 1.0 / (1.0 + expf(-1.0 * temp));
}