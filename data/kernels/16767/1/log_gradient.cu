#include "hip/hip_runtime.h"
#include "includes.h"

//FILE IO RELATED
//max number of lines in the training dataset
#define MAX_ROWS_TRAINING 16896
// max number of columns/features in the training dataset
#define MAX_COLUMNS_TRAINING 26
// max number of rows in the testing dataset
#define MAX_ROWS_TESTING 4096
// max number of columns in the testing data
#define MAX_COLUMNS_TESTING 26
//max number of characters/line
#define MAX_CHAR 300

__constant__ int features = 26;
__constant__ int num_rows = 16896;

long mem_cpy_time = 0;
long beta_cpy_time = 0;

// parallelized across the rows

// parallelized across the features

__global__ void log_gradient(float* log_func_v,  float* gradient, float* betas, float* data, int* yvec) {
// the logistic function itself has been pulled out
int feature_index = blockIdx.x * blockDim.x + threadIdx.x;
float temp = 0.0f;
for(int i = 0; i < num_rows; i++) {
float sub = log_func_v[i] - yvec[i];
float accessed_data = data[(i * features) + feature_index];
temp += sub * accessed_data;
}
gradient[feature_index] = temp;
}