#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void norm_components(float* N, int npix, float* norm) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npix) {
norm[i] = fmaxf(1e-10, sqrtf(N[i] * N[i] + N[npix + i] * N[npix + i] + N[npix * 2 + i] * N[npix * 2 + i]));
}
}