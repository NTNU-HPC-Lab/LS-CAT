#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squared_difference(float *x, float *y, int len) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < len) {
x[i] = (x[i] - y[i])*(x[i] - y[i]);
}
}