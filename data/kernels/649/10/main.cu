#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "calculate_A_ch_1_2.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *rho = NULL;
hipMalloc(&rho, XSIZE*YSIZE);
float *dz = NULL;
hipMalloc(&dz, XSIZE*YSIZE);
float *s_a = NULL;
hipMalloc(&s_a, XSIZE*YSIZE);
float *xx_or_yy = NULL;
hipMalloc(&xx_or_yy, XSIZE*YSIZE);
float *s_b = NULL;
hipMalloc(&s_b, XSIZE*YSIZE);
float K = 1;
int npix = 1;
int nchannels = 1;
int nimages = 1;
float *A_ch = NULL;
hipMalloc(&A_ch, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_A_ch_1_2<<<gridBlock,threadBlock>>>(rho,dz,s_a,xx_or_yy,s_b,K,npix,nchannels,nimages,A_ch);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
calculate_A_ch_1_2<<<gridBlock,threadBlock>>>(rho,dz,s_a,xx_or_yy,s_b,K,npix,nchannels,nimages,A_ch);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
calculate_A_ch_1_2<<<gridBlock,threadBlock>>>(rho,dz,s_a,xx_or_yy,s_b,K,npix,nchannels,nimages,A_ch);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}