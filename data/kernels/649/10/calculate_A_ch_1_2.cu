#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_A_ch_1_2(float* rho, float* dz, float* s_a, float* xx_or_yy, float* s_b, float K, int npix, int nchannels, int nimages, float* A_ch) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int c = blockIdx.z*blockDim.z + threadIdx.z;
if (i < npix && j < nimages) {
A_ch[c*npix*nimages + j*npix + i] = (rho[c*npix + i] / dz[i])*(K*s_a[c * nimages * 3 + j] - xx_or_yy[i] * s_b[c * nimages * 3 + j]);
}
}