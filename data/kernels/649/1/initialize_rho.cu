#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize_rho(float* rho, int size_c, int nc) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int c = blockIdx.y*blockDim.y + threadIdx.y;
if (i < size_c && c < nc) {
rho[c*(size_c)+i] = 0.5f;
}
}