#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_A_ch_3(float* rho, float* dz, float* s_a, int npix, int nchannels, int nimages, float* A_ch) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int c = blockIdx.z*blockDim.z + threadIdx.z;
if (i < npix && j < nimages) {
A_ch[c*npix*nimages + j*npix + i] = (rho[c*npix + i] / dz[i])*(s_a[c * nimages * 3 + j]);
}
}