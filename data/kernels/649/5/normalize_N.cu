#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_N(float* N, float* norm, int npix_per_component) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int c = blockIdx.y*blockDim.y + threadIdx.y;
if (i < npix_per_component) {
N[c*npix_per_component + i] = N[c*npix_per_component + i] / norm[i];
}
}