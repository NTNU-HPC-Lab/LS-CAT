#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, int npix, float* N3) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npix) {
N3[i] = -z[i] - (xx[i]) * zx[i] - (yy[i]) * zy[i];
N3[npix + i] = 1;
}
}