#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_constant(int* arr, int k, int arr_size) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < arr_size) {
arr[i] += k;
}
}