#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_AT_expansion(float* A, int* rowind, int* colind, float* val, int npix, int nimages) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npix*nimages) {
colind[i] = i / nimages + (i % nimages)*npix;
rowind[i] = i / nimages;
val[i] = A[colind[i]];
}
}