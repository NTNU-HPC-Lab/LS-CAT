#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_B_for_depth(float* B, float* rho, float* Ns, int npix, int nchannels, int nimages) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int c = blockIdx.y*blockDim.y + threadIdx.y;
if (i < npix*nimages) {
B[c*npix*nimages + i] -= rho[c*npix + i%npix] * Ns[c*npix*nimages + i];
}
}