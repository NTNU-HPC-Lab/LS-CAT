#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_A_expansion(float* A, int* rowind, int* colind, float* val, int npix, int nimages) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npix*nimages) {
rowind[i] = i;
colind[i] = i % npix;
val[i] = A[i];
}
}