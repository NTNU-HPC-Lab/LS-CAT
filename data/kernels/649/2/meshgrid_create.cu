#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void meshgrid_create(float* xx, float* yy, int w, int h, float K02, float K12) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
if (i < h && j < w) {
xx[j*h + i] = j - K02;
yy[j*h + i] = i - K12;
}
}