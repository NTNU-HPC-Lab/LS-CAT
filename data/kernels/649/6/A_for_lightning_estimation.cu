#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void A_for_lightning_estimation(float* rho, float* N, int npix, float* A) {
int i = blockIdx.x*blockDim.x + threadIdx.x; // pixel index
int c = blockIdx.y*blockDim.y + threadIdx.y; // channel index
int h = blockIdx.z*blockDim.z + threadIdx.z; // harmonic index
if (i < npix) {
A[c*npix * 4 + h*npix + i] = rho[c*npix + i] * N[h*npix + i];
}
}