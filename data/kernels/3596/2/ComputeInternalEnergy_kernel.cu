#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeInternalEnergy_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *Eneint, float *Bx, float *By, float *Bz, int size)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igrid = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;


if (igrid >= size)
return;

// compute internal energy
Eneint[igrid] = Etot[igrid] - 0.5*(Vx[igrid]*Vx[igrid] + Vy[igrid]*Vy[igrid] + Vz[igrid]*Vz[igrid]) -
0.5*(Bx[igrid]*Bx[igrid] + By[igrid]*By[igrid] + Bz[igrid]*Bz[igrid])/Rho[igrid];

}