#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HydroComputedUz_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size, int dim0, int dim1, int dim2)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igridz = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;

if (igridz < 2 || igridz > size - 3)
return;

int j = igridz / (dim0*dim2);
int i = (igridz - j*dim0*dim2) / dim2;
int k = igridz - j*dim0*dim2 - i*dim2;
int igrid = i + (j + k*dim1) * dim0;

int igridzp1 = igridz + 1;
j = igridzp1 / (dim0*dim2);
i = (igridzp1 - j*dim0*dim2) / dim2;
k = igridzp1 - j*dim0*dim2 - i*dim2;
int igridp1 = i + (j + k*dim1) * dim0;

dUD  [igrid] += (FluxD  [igrid] - FluxD  [igridp1])*dtdx;
dUS1 [igrid] += (FluxS1 [igrid] - FluxS1 [igridp1])*dtdx;
dUS2 [igrid] += (FluxS2 [igrid] - FluxS2 [igridp1])*dtdx;
dUS3 [igrid] += (FluxS3 [igrid] - FluxS3 [igridp1])*dtdx;
dUTau[igrid] += (FluxTau[igrid] - FluxTau[igridp1])*dtdx;

}