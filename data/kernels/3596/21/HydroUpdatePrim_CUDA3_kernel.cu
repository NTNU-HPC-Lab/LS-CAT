#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HydroUpdatePrim_CUDA3_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dt, int size)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igrid = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;

if (igrid < 2 || igrid > size - 3)
return;

float D, S1, S2, S3, Tau;
D   = Rho[igrid];
S1  = D*Vx[igrid];
S2  = D*Vy[igrid];
S3  = D*Vz[igrid];
Tau = D*Etot[igrid];

D   += dUD[igrid];
S1  += dUS1[igrid];
S2  += dUS2[igrid];
S3  += dUS3[igrid];
Tau += dUTau[igrid];

Rho[igrid] = D;
Vx[igrid] = S1/D;
Vy[igrid] = S2/D;
Vz[igrid] = S3/D;
Etot[igrid] = Tau/D;

}