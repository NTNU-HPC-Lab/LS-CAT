#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "MHDUpdatePrim_CUDA3_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *Rho = NULL;
hipMalloc(&Rho, XSIZE*YSIZE);
float *Vx = NULL;
hipMalloc(&Vx, XSIZE*YSIZE);
float *Vy = NULL;
hipMalloc(&Vy, XSIZE*YSIZE);
float *Vz = NULL;
hipMalloc(&Vz, XSIZE*YSIZE);
float *Etot = NULL;
hipMalloc(&Etot, XSIZE*YSIZE);
float *Bx = NULL;
hipMalloc(&Bx, XSIZE*YSIZE);
float *By = NULL;
hipMalloc(&By, XSIZE*YSIZE);
float *Bz = NULL;
hipMalloc(&Bz, XSIZE*YSIZE);
float *Phi = NULL;
hipMalloc(&Phi, XSIZE*YSIZE);
float *dUD = NULL;
hipMalloc(&dUD, XSIZE*YSIZE);
float *dUS1 = NULL;
hipMalloc(&dUS1, XSIZE*YSIZE);
float *dUS2 = NULL;
hipMalloc(&dUS2, XSIZE*YSIZE);
float *dUS3 = NULL;
hipMalloc(&dUS3, XSIZE*YSIZE);
float *dUTau = NULL;
hipMalloc(&dUTau, XSIZE*YSIZE);
float *dUBx = NULL;
hipMalloc(&dUBx, XSIZE*YSIZE);
float *dUBy = NULL;
hipMalloc(&dUBy, XSIZE*YSIZE);
float *dUBz = NULL;
hipMalloc(&dUBz, XSIZE*YSIZE);
float *dUPhi = NULL;
hipMalloc(&dUPhi, XSIZE*YSIZE);
float dt = 1;
float C_h = 1;
float C_p = 1;
int size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
MHDUpdatePrim_CUDA3_kernel<<<gridBlock,threadBlock>>>(Rho,Vx,Vy,Vz,Etot,Bx,By,Bz,Phi,dUD,dUS1,dUS2,dUS3,dUTau,dUBx,dUBy,dUBz,dUPhi,dt,C_h,C_p,size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
MHDUpdatePrim_CUDA3_kernel<<<gridBlock,threadBlock>>>(Rho,Vx,Vy,Vz,Etot,Bx,By,Bz,Phi,dUD,dUS1,dUS2,dUS3,dUTau,dUBx,dUBy,dUBz,dUPhi,dt,C_h,C_p,size);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
MHDUpdatePrim_CUDA3_kernel<<<gridBlock,threadBlock>>>(Rho,Vx,Vy,Vz,Etot,Bx,By,Bz,Phi,dUD,dUS1,dUS2,dUS3,dUTau,dUBx,dUBy,dUBz,dUPhi,dt,C_h,C_p,size);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}