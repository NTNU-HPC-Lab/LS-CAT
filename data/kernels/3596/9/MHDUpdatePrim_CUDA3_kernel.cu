#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MHDUpdatePrim_CUDA3_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *Bx, float *By, float *Bz, float *Phi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dt, float C_h, float C_p, int size)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igrid = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;

if (igrid < 2 || igrid > size - 3)
return;

float D, S1, S2, S3, Tau;
D   = Rho[igrid];
S1  = D*Vx[igrid];
S2  = D*Vy[igrid];
S3  = D*Vz[igrid];
Tau = D*Etot[igrid];

D   += dUD[igrid];
S1  += dUS1[igrid];
S2  += dUS2[igrid];
S3  += dUS3[igrid];
Tau += dUTau[igrid];

Rho[igrid] = D;
Vx[igrid] = S1/D;
Vy[igrid] = S2/D;
Vz[igrid] = S3/D;
Etot[igrid] = Tau/D;

Bx[igrid] += dUBx[igrid];
By[igrid] += dUBy[igrid];
Bz[igrid] += dUBz[igrid];
Phi[igrid] += dUPhi[igrid];
Phi[igrid] *= expf(-dt*(C_h/C_p)*(C_h/C_p));
}