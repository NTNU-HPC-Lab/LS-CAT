#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HydroComputedUx_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igrid = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;

if (igrid < 2 || igrid > size - 3)
return;

int igridp1 = igrid + 1;
dUD  [igrid] = (FluxD  [igrid] - FluxD  [igridp1])*dtdx;
dUS1 [igrid] = (FluxS1 [igrid] - FluxS1 [igridp1])*dtdx;
dUS2 [igrid] = (FluxS2 [igrid] - FluxS2 [igridp1])*dtdx;
dUS3 [igrid] = (FluxS3 [igrid] - FluxS3 [igridp1])*dtdx;
dUTau[igrid] = (FluxTau[igrid] - FluxTau[igridp1])*dtdx;

}