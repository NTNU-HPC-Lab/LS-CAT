#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MHDComputedUx_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *FluxBx, float *FluxBy, float *FluxBz, float *FluxPhi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dtdx, int size)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igrid = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;


if (igrid < 2 || igrid > size - 3)
return;

int igridp1 = igrid + 1;
dUD  [igrid] = (FluxD  [igrid] - FluxD  [igridp1])*dtdx;
dUS1 [igrid] = (FluxS1 [igrid] - FluxS1 [igridp1])*dtdx;
dUS2 [igrid] = (FluxS2 [igrid] - FluxS2 [igridp1])*dtdx;
dUS3 [igrid] = (FluxS3 [igrid] - FluxS3 [igridp1])*dtdx;
dUTau[igrid] = (FluxTau[igrid] - FluxTau[igridp1])*dtdx;
dUBx [igrid] = (FluxBx [igrid] - FluxBx [igridp1])*dtdx;
dUBy [igrid] = (FluxBy [igrid] - FluxBy [igridp1])*dtdx;
dUBz [igrid] = (FluxBz [igrid] - FluxBz [igridp1])*dtdx;
dUPhi[igrid] = (FluxPhi[igrid] - FluxPhi[igridp1])*dtdx;

}