#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HydroComputedUy_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size, int dim0, int dim1, int dim2)
{
// get thread and block index
const long tx = threadIdx.x;
const long bx = blockIdx.x;
const long by = blockIdx.y;

int igridy = tx + bx*CUDA_BLOCK_SIZE + by*CUDA_BLOCK_SIZE*CUDA_GRID_SIZE;

if (igridy < 2 || igridy > size - 3)
return;

int k = igridy/(dim0*dim1);
int i = (igridy - k*dim0*dim1)/dim1;
int j = igridy - k*dim0*dim1 - i*dim1;
int igrid = i + (j + k*dim1) * dim0;

int igridyp1 = igridy + 1;
k = igridyp1/(dim0*dim1);
i = (igridyp1 - k*dim0*dim1)/dim1;
j = igridyp1 - k*dim0*dim1 - i*dim1;
int igridp1 = i + (j + k*dim1) * dim0;


dUD  [igrid] += (FluxD  [igrid] - FluxD  [igridp1])*dtdx;
dUS1 [igrid] += (FluxS1 [igrid] - FluxS1 [igridp1])*dtdx;
dUS2 [igrid] += (FluxS2 [igrid] - FluxS2 [igridp1])*dtdx;
dUS3 [igrid] += (FluxS3 [igrid] - FluxS3 [igridp1])*dtdx;
dUTau[igrid] += (FluxTau[igrid] - FluxTau[igridp1])*dtdx;

}