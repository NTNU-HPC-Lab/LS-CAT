#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexSelect( float *tensor, float *src, long* src_stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim )
{
int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

long flat_size = tensor_size / idx_size;

if (thread_idx < flat_size)
{
long coeff = 0;
for (int i=0; i<idx_size; i++)
{
int leftover = thread_idx;
int targetIdx = 0;
int srcIdx = 0;
for (int d=0; d<src_nDim; d++)
{
if (d < dim)
{
long stride_d = src_stride[d] / size_dim;
coeff = leftover / stride_d;
leftover -= coeff * stride_d;
targetIdx += coeff * stride_d * idx_size;
srcIdx += coeff * src_stride[d];
}
else if (d > dim)
{
coeff = leftover / src_stride[d];
leftover -= coeff * src_stride[d];
targetIdx += coeff * src_stride[d];
srcIdx += coeff * src_stride[d];
}
}
tensor[targetIdx + i*src_stride[dim]] = src[srcIdx + ((int)(index[i])-1)*src_stride[dim]];
}
}
}