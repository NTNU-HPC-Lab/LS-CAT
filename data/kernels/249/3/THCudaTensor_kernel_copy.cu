#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_copy(float *dst, long *dst_sz, long *dst_st, int dst_dim, float *src, long *src_sz, long *src_st, int src_dim, long n_elem, long innerdim)
{
long k = (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x)*blockDim.y + threadIdx.y;

long i_start = threadIdx.x * src_st[src_dim-1];
long i_step = blockDim.x * src_st[src_dim-1];

long o_start = threadIdx.x * dst_st[dst_dim-1];
long o_step = blockDim.x * dst_st[dst_dim-1];
long o_end = innerdim * dst_st[dst_dim-1];

if ( ((k+1) * innerdim) <= n_elem) // too safe
{
long dst_idx = 0;
long dst_rest = k * innerdim;
for(int dim = 0; dim < dst_dim; dim++)
{
dst_idx += (dst_rest/dst_sz[dim])*dst_st[dim];
dst_rest = dst_rest % dst_sz[dim];
}

long src_idx = 0;
long src_rest = k * innerdim;
for(int dim = 0; dim < src_dim; dim++)
{
src_idx += (src_rest/src_sz[dim])*src_st[dim];
src_rest = src_rest % src_sz[dim];
}

for (int i=i_start, o=o_start; o<o_end; i+=i_step, o+=o_step) {
dst[dst_idx + o] = src[src_idx + i];
}
}
}