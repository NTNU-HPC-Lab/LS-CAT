#include "hip/hip_runtime.h"
#include "includes.h"


#define _SIZE_ 1000000

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

*/


__global__ void addLoopGPU(int* a, int* b, int* c)
{
int tid = blockIdx.x;
if (tid < 64)
c[tid] = abs(powf(b[tid], 2) - powf(b[tid], 2));
}