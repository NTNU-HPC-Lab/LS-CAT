#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void post_scan(float* in, float* add, int len) {
unsigned int t = threadIdx.x;
unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

if (blockIdx.x) {
if (start + t < len) in[start + t] += add[blockIdx.x - 1];
if (start + BLOCK_SIZE + t < len) in[start + BLOCK_SIZE + t] += add[blockIdx.x - 1];
}
}