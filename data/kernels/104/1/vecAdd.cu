#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
register int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < len) out[i] = in1[i] + in2[i];
}