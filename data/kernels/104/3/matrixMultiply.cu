#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if ((row < numCRows) && (col < numCColumns)) {
float value = 0;
#pragma unroll
for (int k = 0; k < numAColumns; ++k)
value += A[row * numAColumns + k] * B[k * numBColumns + col];
C[row * numCColumns + col] = value;
}
}