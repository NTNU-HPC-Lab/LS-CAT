#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyTiled(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;
unsigned int col = blockIdx.x * TILE_WIDTH + tx;
unsigned int row = blockIdx.y * TILE_WIDTH + ty;
float acc = 0;

for (int t = 0; t < (numAColumns-1)/TILE_WIDTH + 1; ++t) {
unsigned int ATilePitch = t * TILE_WIDTH + tx;
unsigned int BTilePitch = t * TILE_WIDTH + ty;

if (row < numARows && ATilePitch < numAColumns)
ds_A[ty][tx] = A[row * numAColumns + ATilePitch];
else
ds_A[ty][tx] = 0;

if (col < numBColumns && BTilePitch < numBRows)
ds_B[ty][tx] = B[BTilePitch * numBColumns + col];
else
ds_B[ty][tx] = 0;

__syncthreads();
#pragma unroll
for (int k = 0; k < TILE_WIDTH; ++k) acc += ds_A[ty][k] * ds_B[k][tx];
__syncthreads();
}

if (row < numCRows && col < numCColumns) C[row * numCColumns + col] = acc;
}