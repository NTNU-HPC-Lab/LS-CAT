#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
int n,N;
int offset,soffset;

// since we are using multiple threads per blocks as well as multiple blocks
int vidxb = 4*(blockIdx.x * blockDim.x) + threadIdx.x;
//int basevert = 4*(blockIdx.x * blockDim.x);

int vidx; //,tab;
float4 nbd,td;

// create a cache for 4 elements per block (4*BLOCK_SIZE elements)
__shared__ float4 SI[4*BLOCK_SIZE_AVGG];

int bidx = 4*threadIdx.x;
// this means we have 128 neighboring vertices cached
for (vidx=vidxb; vidx<vidxb+4*BLOCK_SIZE_AVGG; vidx+=BLOCK_SIZE_AVGG)
{
if (vidx < nVertices)
{
SI[bidx] = D[vidx];
bidx++;
}
}

__syncthreads();

bidx = 4*threadIdx.x;
// preload the current BLOCK_SIZE vertices
for (vidx=vidxb; vidx<vidxb+4*BLOCK_SIZE_AVGG; vidx+=BLOCK_SIZE_AVGG)
{
if (vidx < nVertices)
{

offset = NBOFFSETS[ vidx ];
N = nNeighbors[ vidx ];

td = SI[bidx++];

for (n = 0; n < N; n++)
{
soffset = NEIGHBOR[offset+n];
/*
tab = soffset - basevert;
if(tab > 0 && tab < 4*BLOCK_SIZE)
nbd = SI[tab];
else
*/
nbd = D[soffset];

td.x += nbd.x;
td.y += nbd.y;
td.z += nbd.z;
}

td.x /= (float)(N+1);
td.y /= (float)(N+1);
td.z /= (float)(N+1);

TD[vidx] = td;
}
}
}