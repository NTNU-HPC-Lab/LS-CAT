#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void run_reduction(int *con, int *blockCon,int* ActiveList, int nActiveBlock, int* blockSizes)
{
int list_idx = blockIdx.y*gridDim.x + blockIdx.x;


if(list_idx < nActiveBlock)
{
int block_idx = ActiveList[list_idx];

__shared__ int s_conv[REDUCTIONSHARESIZE];


uint base_addr = block_idx*blockDim.x*2;   // *2 because there are only half block size number of thread
uint tx = threadIdx.x;


s_conv[tx] = con[base_addr + tx];
s_conv[tx + blockDim.x] = con[base_addr + tx + blockDim.x];

__syncthreads();

for(uint i=blockDim.x; i>0; i/=2)
{
if(tx < i)
{
bool b1, b2;
b1 = s_conv[tx];
b2 = s_conv[tx+i];
s_conv[tx] = (b1 && b2) ? 1 : 0 ;
}
__syncthreads();
}

if(tx == 0)
{
blockCon[block_idx] = s_conv[0]; // active list is negation of tile convergence (active = not converged)
}
}
}