#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_entropy_atomic(float *float_image_in, float *entropy_out, int blk_size) {
//calculate entropy of a block through a single thread
__shared__ float sum;
if (threadIdx.x == 0 && threadIdx.y == 0) {
sum = 0.0;
}
__syncthreads();
int blocksize = blk_size*blk_size;
//vertical offset to get to beginning of own block
int v_offset_to_blkrow = gridDim.x*blockDim.x*blockDim.y*blockIdx.y;
int v_offset_to_pixrow = blockDim.x*gridDim.x*threadIdx.y;
int h_offset = blockDim.x*blockIdx.x + threadIdx.x;
int idx = v_offset_to_blkrow + v_offset_to_pixrow + h_offset; //idx of top left corner of the block
int out_idx = blockIdx.y*gridDim.x + blockIdx.x;
//normalize image
float_image_in[idx] = float_image_in[idx] * float_image_in[idx] / (blocksize);
atomicAdd(&sum, float_image_in[idx]);
__syncthreads();
__shared__ float entropy;
if (threadIdx.x == 0 && threadIdx.y == 0) {
entropy = 0.0;
}
__syncthreads();
float_image_in[idx] = float_image_in[idx] / sum;
//shannon entropy
atomicAdd(&entropy, -float_image_in[idx] * log2(float_image_in[idx]));
__syncthreads();
//printf("%f\n", sum2);
if (threadIdx.x == 0 && threadIdx.y == 0) {
entropy_out[out_idx] = entropy;
}
}