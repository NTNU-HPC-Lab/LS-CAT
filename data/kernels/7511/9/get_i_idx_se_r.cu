#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_i_idx_se_r(const int nloc, const int * ilist, int * i_idx)
{
const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
if(idy >= nloc) {
return;
}
i_idx[ilist[idy]] = idy;
}