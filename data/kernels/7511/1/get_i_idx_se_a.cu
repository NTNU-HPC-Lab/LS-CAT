#include "hip/hip_runtime.h"
#include "includes.h"
/* Copyright 2015 The TensorFlow Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/
#define EIGEN_USE_GPU

#ifdef HIGH_PREC
typedef double  VALUETYPE;
#else
typedef float   VALUETYPE;
#endif

typedef unsigned long long int_64;

__global__ void get_i_idx_se_a(const int nloc, const int * ilist, int * i_idx)
{
const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
if(idy >= nloc) {
return;
}
i_idx[ilist[idy]] = idy;
}