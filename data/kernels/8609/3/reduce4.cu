#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL
__global__ void reduce4(int *g_idata, int *g_odata, int g_size)
{
__shared__ int sdata[blocksize];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
__syncthreads();
// do reduction in shared mem
for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
{
if (tid < s)
{
sdata[tid] += sdata[tid + s];
}

__syncthreads();
}
// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}