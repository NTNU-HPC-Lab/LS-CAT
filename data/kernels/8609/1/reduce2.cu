#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL
__global__ void reduce2(int *g_idata, int *g_odata, int g_size)
{
__shared__ int sdata[blocksize];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = g_idata[i];
__syncthreads();
// do reduction in shared mem
for (unsigned int s = 1; s < blockDim.x; s *= 2)
{
int index = 2 * s*tid;

if (index < blockDim.x)
{
sdata[index] += sdata[index + s];
}

__syncthreads();
}
// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}