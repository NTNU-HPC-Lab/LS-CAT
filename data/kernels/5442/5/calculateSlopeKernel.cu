#include "hip/hip_runtime.h"
#include "includes.h"
///////////////////////////////////////////////////////////////////////////////

//Round a / b to nearest higher integer value
__global__ void calculateSlopeKernel(float* h, float2 *slopeOut, unsigned int width, unsigned int height){
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
unsigned int i = y*width+x;

float2 slope = make_float2(0.0f, 0.0f);
if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
slope.x = h[i+1] - h[i-1];
slope.y = h[i+width] - h[i-width];
}
slopeOut[i] = slope;
}