#include "hip/hip_runtime.h"
#include "includes.h"
///////////////////////////////////////////////////////////////////////////////

//Round a / b to nearest higher integer value
__global__ void updateHeightmapKernel(float*  heightMap, float2* ht, unsigned int width){
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
unsigned int i = y*width+x;

float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;
heightMap[i] = ht[i].x * sign_correction;
}