#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void computearray_size(int* block_cntr_array,int *finalsize,int *orig_number_of_char)
{
*finalsize = 0;
for(int i=0;i<*orig_number_of_char;i++)
{
(*finalsize)=(*finalsize) + block_cntr_array[i];
}

}
__device__ int char_huffman_table_gpu[MAX_CHAR][MAX_CHAR-1];  //To write the output from compression in GPU  //char *compressedfile_array=0;  bool *compressedfile_array=0;  bool *finalcompressed_array=0;  // To keep track of how many characters each block wrote  int *block_cntr_array=0; int *block_cntr_array_check=0; int *d_last_byte_padding=0;  int *finalsize=0; int *orig_number_of_char=0; int *huffman_check = (int *)malloc((MAX_CHAR)*(MAX_CHAR-1) *sizeof(int));

bool *d_bool = 0;

bool *h_bool = 0;



__global__ void final_compression(int *block_cntr_array,bool *compressedfile_array,bool *finalcompressed_array,int number_of_char)
//__device__ void final_compression(int *block_cntr_array,bool *compressedfile_array,bool *finalcompressed_array)
{
int index_blocks=blockIdx.x*blockDim.x+threadIdx.x;
int index_file=(blockIdx.x*blockDim.x+threadIdx.x)*255;
int final_index=0;

if(index_blocks < number_of_char)
{
for(int i=0;i<index_blocks;i++)
{
final_index = final_index+ block_cntr_array[i];
}
for(int i=0;i<block_cntr_array[index_blocks];i++)
{
finalcompressed_array[final_index+i]=compressedfile_array[index_file+i];
}

}
}
__global__ void compress_file_gpu(unsigned char *d_input,bool *compressedfile_array,int *char_huffman_table2,int *block_cntr_array,int* d_last_byte_padding,int *finalsize,int *orig_number_of_char,int number_of_char)
{
//int write_counter=0,
int block_counter=0;	//how many bits have been written in specific byte
unsigned char input_char;
//unsigned char output_char = 0x0;
//unsigned char end_of_file = 255;
//unsigned char mask = 0x01; //00000001;
int index_file=(blockIdx.x*blockDim.x+threadIdx.x)*255;
int index_blocks=blockIdx.x*blockDim.x+threadIdx.x;

if(index_blocks < number_of_char)
{
//for(int i=0;i<MAX_CHAR;i++)
//{
//int *row = (int*)((char*)char_huffman_table2 + i * pitch);
//for (int c = 0; c < MAX_CHAR-1; ++c) {
//   char_huffman_table_gpu[i][c] = row[c];
//}
//}

input_char = d_input[index_blocks];
for(int i = 0 ; i < (MAX_CHAR - 1) ; i++)
{
if(char_huffman_table2[input_char*255+i] == 0)			//detect if current character on particular position has 0 or 1
{
//output_char = output_char << 1;					//if 0 then shift bits one position to left (last bit after shifting is 0)
compressedfile_array[index_file+i] = false;
//write_counter++;
block_counter++;
}
else if(char_huffman_table2[input_char*255+i] == 1)
{
//output_char = output_char << 1;					//if 1 then shift bits one position to left...
//output_char = output_char | mask;				//...and last bit change to: 1
//write_counter++;
compressedfile_array[index_file+i] = true;
block_counter++;
}
else //-1
{
/*if(input_char == end_of_file)					//if EOF is detected then write current result to file
{
if(write_counter != 0)
{
output_char = output_char << (8-write_counter);
compressedfile_array[index_file]=output_char;
output_char = 0x0;
}
else	//write_counter == 0
{
compressedfile_array[index_file]=output_char;
}
}*/

break;
}

/*if(write_counter == 8)								//if result achieved 8 (size of char) then write it to compressed_file
{
compressedfile_array[index_file]=output_char;
output_char = 0x0;
write_counter = 0;
}*/
}

block_cntr_array[index_blocks]=block_counter;
//*d_last_byte_padding = write_counter;							//to decompress file we have to know how many bits in last byte have been written
//update_config(write_counter); //TODO to zakomentowac przy ostatecznych pomiarach

computearray_size(block_cntr_array,finalsize,orig_number_of_char);
//final_compression(block_cntr_array,compressedfile_array,finalcompressed_array);

}

}