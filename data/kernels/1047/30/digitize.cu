#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void digitize(float* idat, uint8_t* udat, size_t n)
{
for (
int i = threadIdx.x + blockIdx.x*blockDim.x;
i < n;
i += blockDim.x*gridDim.x)
{
// add an extra 2 here for overhead in case we make it bright
//float tmp = idat[i]/0.02957/2 + 127.5;
// this normalization appears to be more consistent with the VLITE
// digitizers, which have a mean of 128
float tmp = idat[i]/0.02957/2 + 128.5;
if (tmp <= 0)
udat[i] = 0;
else if (tmp >= 255)
udat[i] = 255;
else
udat[i] = (uint8_t) tmp;
}
}