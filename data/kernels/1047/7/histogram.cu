#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram ( unsigned char *utime, unsigned int* histo, size_t n)
{
__shared__ unsigned int lhisto[512];
lhisto[threadIdx.x] = 0;
__syncthreads ();

int i = threadIdx.x + blockIdx.x*blockDim.x;
for (; i < n/2; i += blockDim.x*gridDim.x)
atomicAdd (lhisto+utime[i], 1);
for (; i < n; i += blockDim.x*gridDim.x)
atomicAdd ((lhisto+256)+utime[i], 1);
__syncthreads ();

// MUST run with 512 threads for this global accumulation to work
atomicAdd ( histo+threadIdx.x, lhisto[threadIdx.x]);
}