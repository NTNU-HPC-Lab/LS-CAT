#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;


__device__ void swap(int *a, int *b) {
int temp = *a;
*a = *b;
*b = temp;
}
__global__ void sort(int *d_arr, int n, bool isEven) {
int i;
if (isEven) {
i = threadIdx.x * 2;
} else {
i = threadIdx.x * 2 + 1;
}

if (i < n -1) {
if (d_arr[i] > d_arr[i + 1]) {
swap(&d_arr[i], &d_arr[i + 1]);
}
}
}