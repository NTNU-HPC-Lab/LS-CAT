#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPU_vector_add(int* left, int* right, int* result)
{
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
result[idx] = left[idx] + right[idx];
}