#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void edgeDetect(unsigned char* device_input_data, unsigned char* device_output_data, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

//check bounds
if (x < 1 || x > width - 1 || y > height - 1 || y < 1)
return;

//for horizontal lines
const int fmat_x[3][3] = {
{-1, 0, 1},
{-2, 0, 2},
{-1, 0, 1}
};
// for vertical lines
const int fmat_y[3][3]  = {
{-1, -2, -1},
{0,   0,  0},
{1,   2,  1}
};

double G_x = 0;
double G_y = 0;
int G;
//go through rows and cols
for (int i = y - 3 / 2; i < y + 3 - 3 / 2; i++) {
for (int j = x - 3 / 2; j < x + 3 - 3 / 2; j++) {
G_x += (double)(fmat_x[i - y + 3 / 2][x - j + 3 / 2] * device_input_data[i * width + j]);
G_y += (double)(fmat_y[i - y + 3 / 2][x - j + 3 / 2] * device_input_data[i * width + j]);
}
}

G = sqrt(G_x * G_x + G_y * G_y);

if (G < 0)
G = 0;
if (G > 255)
G = 255;

device_output_data[y * width + x] = G;
}