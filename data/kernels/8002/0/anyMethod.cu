#include "hip/hip_runtime.h"
#include "includes.h"

//macro to check return value of the cuda runtime call and exits
//if call failed
__global__ void anyMethod(unsigned char* buff , unsigned char* buffer_out , int w , int h)
{
int x = blockIdx.x * blockDim.x +threadIdx.x ;
int y = blockIdx.y * blockDim.y +threadIdx.y;
int width = w , height = h;

if((x>=0 && x < width) && (y>=0 && y<height))
{
int hx = -buff[width*(y-1) + (x-1)] + buff[width*(y-1)+(x+1)]
-2*buff[width*(y)+(x-1)] + 2* buff[width*(y)+(x+1)]
-buff[width*(y+1)+(x-1)] + buff[width*(y+1)+(x+1)];

int vx = buff[width*(y-1)+(x-1)] +2*buff[width*(y-1)+(x+1)] +buff[width*(y-1)+(x+1)]
-buff[width*(y+1)+(x-1)] -2* buff[width*(y+1)+(x)] - buff[width*(y+1)+(x+1)];
//this is the main part changed to get the sort of tie dye effect for at least
//the first part of the picture
hx = hx*4;
vx = vx/5;

int val = (int)sqrt((float)(hx) * (float)(hx) + (float)(vx) * (float)(vx));

buffer_out[y * width + x] = (unsigned char) val;
}
}