#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_init(int *mapad, int max, int size)
{
/*Identificaciones necesarios*/
/*int IDX_Thread = threadIdx.x;
int IDY_Thread = threadIdx.y;
int IDX_block =	blockIdx.x;
int IDY_block =	blockIdx.y;
int shapeGrid_X = gridDim.x;
int threads_per_block =	blockDim.x * blockDim.y;
int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);*/

int position = blockDim.x * blockDim.y * ((blockIdx.y * gridDim.x)+blockIdx.x)+((threadIdx.y*blockDim.x)+threadIdx.x);

if (position<size) mapad[position] = max;
}