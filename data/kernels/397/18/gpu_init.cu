#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_init(int *mapad, int max, int size){

/*Identificaciones necesarios*/
int IDX_Thread = threadIdx.x;	/*Identificacion del hilo en la dimension*/
int IDY_Thread = threadIdx.y;	/*Identificacion del hilo en la dimension y*/
int IDX_block =	blockIdx.x;	/*Identificacion del bloque en la dimension x*/
int IDY_block = blockIdx.y;	/*Identificacion del bloque en la dimension y */
int shapeGrid_X = gridDim.x; 	/*Numeros del bloques en la dimension */

int threads_per_block =	blockDim.x * blockDim.y; /* Numero de hilos por bloque (1 dimension) */

/*Formula para calcular la posicion*/	//Posicion del vector dependiendo del hilo y del bloque
int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);

//inicializamos
if(position<size) mapad[position] = max;
}