#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Computación Paralela (curso 1516)
*
* Alberto Gil
* Guillermo Cebrian
*/


// Includes generales


// Include para las utilidades de computación paralela

/**
* Estructura antena
*/
typedef struct {
int y;
int x;
} Antena;

/**
* Macro para acceder a las posiciones del mapa
*/

#define m(y,x) mapa[ (y * cols) + x ]

/**
* Definimos el tamaño de bloque
*/
#define TAMBLOCK 128

/* Funcion que inicializa la matriz al valor maximo */


/* Actualiza el mapa despues de colocar una antena*/




/**
* Función de ayuda para imprimir el mapa
*/

__global__ void gpu_init(int *mapad, int max, int size){

/*Identificaciones necesarios*/
int IDX_Thread = threadIdx.x;	/*Identificacion del hilo en la dimension*/
int IDY_Thread = threadIdx.y;	/*Identificacion del hilo en la dimension y*/
int IDX_block =	blockIdx.x;	/*Identificacion del bloque en la dimension x*/
int IDY_block = blockIdx.y;	/*Identificacion del bloque en la dimension y */
int shapeGrid_X = gridDim.x; 	/*Numeros del bloques en la dimension */

int threads_per_block =	blockDim.x * blockDim.y; /* Numero de hilos por bloque (1 dimension) */

/*Formula para calcular la posicion*/	//Posicion del vector dependiendo del hilo y del bloque
int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);

//inicializamos
if(position<size) mapad[position] = max;
}