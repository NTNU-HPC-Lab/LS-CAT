#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_reduce(int *c, int size)
{
/*Identificaciones necesarios*/
int IDX_Thread = threadIdx.x;
int IDY_Thread = threadIdx.y;
int IDX_block =	blockIdx.x;
int IDY_block =	blockIdx.y;
int shapeGrid_X = gridDim.x;
int threads_per_block =	blockDim.x * blockDim.y;
int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);
if(position<size){
if(size%2 != 0)
{
if(c[position]<c[size-1])
{
c[position]=c[size-1];
}
}else{

if(c[position]<c[position+size/2])
{
c[position]=c[position+size/2];
}
}
}
}