#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Computación Paralela (curso 1516)
*
* Alberto Gil
* Guillermo Cebrian
*/


// Includes generales


// Include para las utilidades de computación paralela

/**
* Estructura antena
*/
typedef struct {
int y;
int x;
} Antena;

/**
* Macro para acceder a las posiciones del mapa
*/

#define m(y,x) mapa[ (y * cols) + x ]

/**
* Definimos el tamaño de bloque
*/
#define TAMBLOCK 128

/* Funcion que inicializa la matriz al valor maximo */


/* Actualiza el mapa despues de colocar una antena*/




/**
* Función de ayuda para imprimir el mapa
*/

__global__ void gpu_reduce(int *c, int size)
{
/*Identificaciones necesarios*/
int IDX_Thread = threadIdx.x;
int IDY_Thread = threadIdx.y;
int IDX_block =	blockIdx.x;
int IDY_block =	blockIdx.y;
int shapeGrid_X = gridDim.x;
int threads_per_block =	blockDim.x * blockDim.y;
int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);


if(position<size){
if(size%2 != 0)
{
if(c[position]<c[size-1])
{
c[position]=c[size-1];
}
}else{

if(c[position]<c[position+size/2])
{
c[position]=c[position+size/2];
}
}
}
}