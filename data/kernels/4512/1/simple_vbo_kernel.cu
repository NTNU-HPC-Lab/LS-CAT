#include "hip/hip_runtime.h"
#include "includes.h"
// Taken from the NVIDIA "2_Graphics\simpleGL" sample:

// A kernel that modifies the z-coordinates of a rectangular
// grid of vertices, based on a time value, so that they
// form an animated sine wave

extern "C"
__global__ void simple_vbo_kernel( float4 *pos, unsigned int width, unsigned int height, float time)
{
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

// calculate uv coordinates
float u = x / (float) width;
float v = y / (float) height;
u = u*2.0f - 1.0f;
v = v*2.0f - 1.0f;

// calculate simple sine wave pattern
float freq = 4.0f;
float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

// write output vertex
pos[y*width+x] = make_float4(u, w, v, 1.0f);
}