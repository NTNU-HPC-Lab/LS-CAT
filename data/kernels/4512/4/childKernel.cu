#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void childKernel(unsigned int parentThreadIndex, float* data)
{
printf("Parent thread index: %d, child thread index: %d\n",
parentThreadIndex, threadIdx.x);
data[threadIdx.x] = parentThreadIndex + 0.1f * threadIdx.x;
}