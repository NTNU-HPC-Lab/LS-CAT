#include "hip/hip_runtime.h"
#include "includes.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
Max dimensions of the world: 6000 x 5500
Parameters: Intel Core i5-2500K 3.30GHz
8GB RAM
NVIDIA GeForce GTX 560 Ti
*/

// includes, system

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#endif

// includes, cuda

// Utilities and timing functions

// CUDA helper functions




__global__ void UpdateGrid(int offX, int offY ,int *i_cells, struct uchar4 *dst, int width, int height, int worldW, int worldH)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x; //position in visible grid
int posX =offX + idx%width;
int posY =offY + idx / width;


int cellIdx = posX*worldW + posY; //calc position of cell in grid
if ( idx < width*height)
{
//assign color
dst[idx].x = i_cells[cellIdx] * 255;
dst[idx].y = i_cells[cellIdx] * 255;
dst[idx].z = i_cells[cellIdx] * 255;
}
}