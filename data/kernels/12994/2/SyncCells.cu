#include "hip/hip_runtime.h"
#include "includes.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
Max dimensions of the world: 6000 x 5500
Parameters: Intel Core i5-2500K 3.30GHz
8GB RAM
NVIDIA GeForce GTX 560 Ti
*/

// includes, system

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#endif

// includes, cuda

// Utilities and timing functions

// CUDA helper functions




__global__ void SyncCells(int *i_cells, int *o_cells, int size)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx >= size)
return;
i_cells[idx] = o_cells[idx];
}