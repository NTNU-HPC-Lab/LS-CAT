#include "hip/hip_runtime.h"
#include "includes.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
Max dimensions of the world: 6000 x 5500
Parameters: Intel Core i5-2500K 3.30GHz
8GB RAM
NVIDIA GeForce GTX 560 Ti
*/

// includes, system

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#endif

// includes, cuda

// Utilities and timing functions

// CUDA helper functions




__device__ int CountAliveCells(int *i_cells, int idx, int width, int height)
{
int alive = 0;

int posY = floorf(idx / width);
int posX = idx % width;

for (int i = -1; i <= 1; i++)
{
for (int j = -1; j <= 1; j++)
{
int currPosX = (posX + i) % width;
int currPosY = (posY + j) % height;

if (currPosX < 0)
{
currPosX = width + currPosX;
}
if (currPosY < 0)
{
currPosY = height + currPosY;
}

int neigh = currPosY * width + currPosX;

if (i == 0 && j == 0)
continue;
if (i_cells[neigh] == 1)
alive++;
}
}
return alive;
}
__global__ void CalcNextGeneration(int *i_cells, int *o_cells, int width, int height)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;

if (idx >= width*height)
return;

int neighCount = CountAliveCells(i_cells, idx, width, height);


if (neighCount == 3 ||
(neighCount == 2 && i_cells[idx] == 1))
o_cells[idx] = 1;
else
o_cells[idx] = 0;

__syncthreads();

}