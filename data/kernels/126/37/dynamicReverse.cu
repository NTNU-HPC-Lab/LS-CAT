#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dynamicReverse(int *d, int n)
{
extern __shared__ int s[];
int t = threadIdx.x;
int tr = n-t-1;
s[t] = d[t];
__syncthreads();
d[t] = s[tr];
}