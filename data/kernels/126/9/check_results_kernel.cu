#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_results_kernel( int n, double correctvalue, double * x )
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n)
{
if ( x[i] != correctvalue )
{
printf("ERROR at index = %d, expected = %f, actual: %f\n",i,correctvalue,x[i]);
}
}
}