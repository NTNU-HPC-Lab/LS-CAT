#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
__shared__ float tile[TILE_DIM][TILE_DIM+1];

int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

__syncthreads();

x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
y = blockIdx.x * TILE_DIM + threadIdx.y;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}