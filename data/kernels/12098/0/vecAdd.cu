#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
c[id] = a[id] * b[id];
}