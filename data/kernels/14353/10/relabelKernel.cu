#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relabelKernel(int *components, int previousLabel, int newLabel, const int colsComponents) {
uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
uint j = (blockIdx.y * blockDim.y) + threadIdx.y;

if (components[i * colsComponents + j] == previousLabel) {
components[i * colsComponents + j] = newLabel;
}
}