#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relabelUnrollKernel(int *components, int previousLabel, int newLabel, const int colsComponents, const int idx, const int frameRows, const int factor) {
uint id_i_child = (blockIdx.x * blockDim.x) + threadIdx.x;
id_i_child = id_i_child + (frameRows * idx);
uint id_j_child = (blockIdx.y * blockDim.y) + threadIdx.y;
id_j_child = (colsComponents / factor) * id_j_child;
uint i = id_i_child;
for (int j = id_j_child; j < (colsComponents / factor); j++) {
if (components[i * colsComponents + j] == previousLabel) {
components[i * colsComponents + j] = newLabel;
}
}
}