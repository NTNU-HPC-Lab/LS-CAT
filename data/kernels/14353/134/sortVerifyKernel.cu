#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortVerifyKernel(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *errNum)
{
uint idx = blockIdx.x * blockDim.x + threadIdx.x;
uint iterator;

#pragma unroll
for (iterator = 0; iterator < THREAD_SIZE; iterator++)
if ((d_SrcKey[d_DstVal[idx*THREAD_SIZE + iterator]] != d_DstKey[idx*THREAD_SIZE + iterator]))
atomicAdd(errNum, 1);
}