#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float_t d_randu(int * seed, int index) {

int M = INT_MAX;
int A = 1103515245;
int C = 12345;
int num = A * seed[index] + C;
seed[index] = num % M;

return fabsf(seed[index] / ((float_t) M));
}
__device__ void cdfCalc(float_t * CDF, float_t * weights, int Nparticles) {
int x;
CDF[0] = weights[0];
for (x = 1; x < Nparticles; x++) {
CDF[x] = weights[x] + CDF[x - 1];
}
}
__global__ void normalize_weights_kernel(float_t * weights, int Nparticles, float_t* partial_sums, float_t * CDF, float_t * u, int * seed) {
int block_id = blockIdx.x;
int i = blockDim.x * block_id + threadIdx.x;
__shared__ float_t u1, sumWeights;

if (0 == threadIdx.x)
sumWeights = partial_sums[0];

__syncthreads();

if (i < Nparticles) {
weights[i] = weights[i] / sumWeights;
}

__syncthreads();

if (i == 0) {
cdfCalc(CDF, weights, Nparticles);
u[0] = (1 / ((float_t) (Nparticles))) * d_randu(seed, i); // do this to allow all threads in all blocks to use the same u1
}

__syncthreads();

if (0 == threadIdx.x)
u1 = u[0];

__syncthreads();

if (i < Nparticles) {
u[i] = u1 + i / ((float_t) (Nparticles));
}
}