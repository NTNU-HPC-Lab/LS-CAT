#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relabel2Kernel(int *components, int previousLabel, int newLabel, const int colsComponents, const int idx, const int frameRows) {
uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
uint j = (blockIdx.y * blockDim.y) + threadIdx.y;
i = i * colsComponents + j;
i = i + (colsComponents * frameRows * idx);
if (components[i] == previousLabel) {
components[i] = newLabel;
}

}