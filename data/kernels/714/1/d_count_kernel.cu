#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_count_kernel(unsigned int * d_pivots, int * r_buckets, int pivotsLength, unsigned int * r_indices, unsigned int * r_sublist, unsigned int * d_in, int itemCount) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < itemCount) {
unsigned int element = d_in[idx];
unsigned int index = pivotsLength/2 - 1;
unsigned int jump = pivotsLength/4;
int pivot = d_pivots[index];
while(jump >= 1) {
index = (element < pivot) ? (index - jump) : (index + jump);
pivot = d_pivots[index];
jump /= 2;
}
index = (element < pivot) ? index : index + 1;
r_sublist[idx] = index;
r_indices[idx] = atomicAdd(&r_buckets[index], 1);
}
}