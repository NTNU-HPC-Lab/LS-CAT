#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_bucketsort(unsigned int * d_in, unsigned int * d_indices, unsigned int * d_sublist, unsigned int * r_outputlist, unsigned int * d_bucketoffsets, int itemCount) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < itemCount) {
int newpos = d_bucketoffsets[d_sublist[idx]] + d_indices[idx];
r_outputlist[newpos] = d_in[idx];
}
}