#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementMulMatrixKernel(double *dev_w, const double *dev_U, const double *dev_V, unsigned int index_row_i, unsigned int index_column_j, unsigned int dim1_U, unsigned int dim1_V)
{
//---------------------------------------------------------------------------------------------------------
// determine indices - row first
//---------------------------------------------------------------------------------------------------------

// 1D
//int idx_k = threadIdx.x;
unsigned int idx_k = blockIdx.x * gridDim.x + threadIdx.x;

// check index range to abort
if (idx_k > dim1_U-1)
return;

unsigned int idx_u_i0 = index_row_i * dim1_U;
unsigned int idx_v_0j = index_column_j;

unsigned int idx_u_ik = idx_u_i0 + idx_k;
unsigned int idx_v_kj = idx_v_0j + idx_k*dim1_V;

//---------------------------------------------------------------------------------------------------------

do
{
//---------------------------------------------------------------------------------------------------------
// access the arrays - row major
//---------------------------------------------------------------------------------------------------------

dev_w[idx_k] = dev_U[idx_u_ik] * dev_V[idx_v_kj];

//---------------------------------------------------------------------------------------------------------
// determine new indices - row first
//---------------------------------------------------------------------------------------------------------

// 1D
//idx_k += blockIdx.x;
idx_k += blockIdx.x * gridDim.x;

idx_u_ik = idx_u_i0 + idx_k;
idx_v_kj = idx_v_0j + idx_k*dim1_V;

} while (idx_k < dim1_U);
//---------------------------------------------------------------------------------------------------------

}