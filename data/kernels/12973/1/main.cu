#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "elementMulMatrixKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *dev_w = NULL;
hipMalloc(&dev_w, XSIZE*YSIZE);
const double *dev_U = NULL;
hipMalloc(&dev_U, XSIZE*YSIZE);
const double *dev_V = NULL;
hipMalloc(&dev_V, XSIZE*YSIZE);
unsigned int index_row_i = 1;
unsigned int index_column_j = 1;
unsigned int dim1_U = 1;
unsigned int dim1_V = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
elementMulMatrixKernel<<<gridBlock,threadBlock>>>(dev_w,dev_U,dev_V,index_row_i,index_column_j,dim1_U,dim1_V);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
elementMulMatrixKernel<<<gridBlock,threadBlock>>>(dev_w,dev_U,dev_V,index_row_i,index_column_j,dim1_U,dim1_V);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
elementMulMatrixKernel<<<gridBlock,threadBlock>>>(dev_w,dev_U,dev_V,index_row_i,index_column_j,dim1_U,dim1_V);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}