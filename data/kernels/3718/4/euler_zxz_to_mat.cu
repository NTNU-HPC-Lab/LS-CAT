#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__global__ void euler_zxz_to_mat(float* afEuler, float* afMat, int iNAngle){
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<iNAngle){
float s1 = sin(afEuler[i * 3 + 0]);
float s2 = sin(afEuler[i * 3 + 1]);
float s3 = sin(afEuler[i * 3 + 2]);
float c1 = cos(afEuler[i * 3 + 0]);
float c2 = cos(afEuler[i * 3 + 1]);
float c3 = cos(afEuler[i * 3 + 2]);
afMat[i * 9 + 0] = c1 * c3 - c2 * s1 * s3;
afMat[i * 9 + 1] = -c1 * s3 - c3 * c2 * s1;
afMat[i * 9 + 2] = s1 * s2;
afMat[i * 9 + 3] = s1 * c3 + c2 * c1 * s3;
afMat[i * 9 + 4] = c1 * c2 * c3 - s1 * s3;
afMat[i * 9 + 5] = -c1 * s2;
afMat[i * 9 + 6] = s3 * s2;
afMat[i * 9 + 7] = s2 * c3;
afMat[i * 9 + 8] = c2;
}
}