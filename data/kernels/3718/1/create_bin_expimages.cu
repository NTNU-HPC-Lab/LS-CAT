#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__global__ void create_bin_expimages(char* acExpDetImages, const int* aiDetStartIdx, const float* afDetInfo,const int iNDet, const int iNRot, const int* aiDetIndex, const int* aiRotN, const int* aiJExp,const int* aiKExp, int const iNPeak){
/*
* create the image matrix
* acExpDetImages: Sigma_i(iNDet*iNRot*iNJ[i]*iNK[i]) , i for each detector, detectors may have different size
* aiDetStartIdx:   index of Detctor start postition in self.acExpDetImages,
* 					e.g. 3 detectors with size 2048x2048, 180 rotations,
* 			 		aiDetStartIdx = [0,180*2048*2048,2*180*2048*2048]
* afDetInfo: iNDet*19, detector information
* iNDet: number of detectors, e.g. 2 or 3;
* iNRot: number of rotations, e.g. 180,720;
* aiDetIndex: len=iNPeak the index of detector, e.g. 0,1 or 2
* aiRotN: aiJExp: aiKExp: len=iNPeak
* iNPeak number of diffraction peaks
* test ?
*/
int i = blockIdx.x*blockDim.x+threadIdx.x;
if(i<iNPeak){
acExpDetImages[aiDetStartIdx[aiDetIndex[i]]
+ aiRotN[i]*int(afDetInfo[0+19*aiDetIndex[i]])*int(afDetInfo[1+19*aiDetIndex[i]])
+ aiKExp[i]*int(afDetInfo[0+19*aiDetIndex[i]]) + aiJExp[i]] = 1;
}
}