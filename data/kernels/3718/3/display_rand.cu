#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__global__ void display_rand(float* afRandom, int iNRand){
int i = blockIdx.x*blockDim.x + threadIdx.x;
printf("=%d=",i);
if (i<iNRand){
printf(" %f ||", afRandom[i]);
}
}