#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__global__ void mat_to_euler_ZXZ(float* afMatIn, float* afEulerOut, int iNAngle){
/*
* transform active rotation matrix to euler angles in ZXZ convention, not right(seems right now)
* afMatIn: iNAngle * 9
* afEulerOut: iNAngle* 3
* TEST PASSED
*/
float threshold = 0.9999999;
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<iNAngle){
if(afMatIn[i * 9 + 8] > threshold){
afEulerOut[i * 3 + 0] = 0;
afEulerOut[i * 3 + 1] = 0;
afEulerOut[i * 3 + 2] = atan2(afMatIn[i*9 + 3], afMatIn[i*9 + 0]);           //  atan2(m[1, 0], m[0, 0])
}
else if(afMatIn[i * 9 + 8] < - threshold){
afEulerOut[i * 3 + 0] = 0;
afEulerOut[i * 3 + 1] = PI;
afEulerOut[i * 3 + 2] = atan2(afMatIn[i*9 + 1], afMatIn[i*9 + 0]);           //  atan2(m[0, 1], m[0, 0])
}
else{
afEulerOut[i * 3 + 0] = atan2(afMatIn[i*9 + 2], - afMatIn[i*9 + 5]);          //  atan2(m[0, 2], -m[1, 2])
afEulerOut[i * 3 + 1] = atan2( sqrt(afMatIn[i*9 + 6] * afMatIn[i*9 + 6]
+ afMatIn[i*9 + 7] * afMatIn[i*9 + 7]),
afMatIn[i*9 + 8]);                             //     atan2(np.sqrt(m[2, 0] ** 2 + m[2, 1] ** 2), m[2, 2])
afEulerOut[i * 3 + 2] = atan2( afMatIn[i*9 + 6], afMatIn[i*9 + 7]);           //   atan2(m[2, 0], m[2, 1])
if(afEulerOut[i * 3 + 0] < 0){
afEulerOut[i * 3 + 0] += 2 * PI;
}
if(afEulerOut[i * 3 + 1] < 0){
afEulerOut[i * 3 + 1] += 2 * PI;
}
if(afEulerOut[i * 3 + 2] < 0){
afEulerOut[i * 3 + 2] += 2 * PI;
}
}
}
}