#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__device__ void d_euler_zxz_to_mat(float* afEuler, float* afMat){
float s1 = sin(afEuler[0]);
float s2 = sin(afEuler[1]);
float s3 = sin(afEuler[2]);
float c1 = cos(afEuler[0]);
float c2 = cos(afEuler[1]);
float c3 = cos(afEuler[2]);
afMat[0] = c1 * c3 - c2 * s1 * s3;
afMat[1] = -c1 * s3 - c3 * c2 * s1;
afMat[2] = s1 * s2;
afMat[3] = s1 * c3 + c2 * c1 * s3;
afMat[4] = c1 * c2 * c3 - s1 * s3;
afMat[5] = -c1 * s2;
afMat[6] = s3 * s2;
afMat[7] = s2 * c3;
afMat[8] = c2;
}
__global__ void rand_mat_neighb_from_euler(float* afEulerIn, float* afMatOut, float* afRand, float fBound){
/* generate random matrix according to the input EulerAngle
* afEulerIn: iNEulerIn * 3, !!!!!!!!!! in radian  !!!!!!!!
* afMatOut: iNNeighbour * iNEulerIn * 9
* afRand:   iNNeighbour * iNEulerIn * 3
* fBound: the range for random angle [-fBound,+fBound]
* iNEulerIn: number of Input Euler angles
* iNNeighbour: number of random angle generated for EACH input
* call:: <<(iNNeighbour,1),(iNEulerIn,1,1)>>
* TEST PASSED
*/
//printf("%f||",fBound);
// keep the original input
float afEulerTmp[3];

afEulerTmp[0] = afEulerIn[threadIdx.x * 3 + 0] + (2 * afRand[blockIdx.x * blockDim.x * 3 + threadIdx.x * 3 + 0] - 1) * fBound;
afEulerTmp[2] = afEulerIn[threadIdx.x * 3 + 2] + (2 * afRand[blockIdx.x * blockDim.x * 3 + threadIdx.x * 3 + 2] - 1) * fBound;
float z = cos(afEulerIn[threadIdx.x * 3 + 1]) +
(afRand[blockIdx.x * blockDim.x * 3 + threadIdx.x * 3 + 1] * 2 - 1) * sin(afEulerIn[threadIdx.x * 3 + 1] * fBound);
if(z>1){
z = 1;
}
else if(z<-1){
z = -1;
}
afEulerTmp[1] = acosf(z);

if(blockIdx.x>0){
d_euler_zxz_to_mat(afEulerTmp, afMatOut + blockIdx.x * blockDim.x * 9 + threadIdx.x * 9);
}
else{
// keep the original input
d_euler_zxz_to_mat(afEulerIn + threadIdx.x * 3, afMatOut + blockIdx.x * blockDim.x * 9 + threadIdx.x * 9);
}
}