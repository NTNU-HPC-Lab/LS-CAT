#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
int iNPixelJ, iNPixelK;
float fPixelJ, fPixelK;
float afCoordOrigin[3];
float afNorm[3];
float afJVector[3];
float afKVector[3];
float fNRot, fAngleStart,fAngleEnd;


} DetInfo;
__device__ void mat3_dot(float* afResult, float* afM0, float* afM1){
/*
* dot product of two 3x3 matrix
*/
for(int i=0;i<3;i++){
for(int j=0;j<3;j++){
afResult[i * 3 + j] = 0;
for(int k=0;k<3;k++){
afResult[i * 3 + j] += afM0[i * 3 + k] * afM1[k * 3 + j];
}
}
}
}
__device__ void mat3_transpose(float* afOut, float* afIn){
/*
* transpose 3x3 matrix
*/
for(int i=0;i<3;i++){
for(int j=0;j<3;j++){
afOut[i * 3 + j] = afIn[j * 3 + i];
}
}
}
__global__ void misorien(float* afMisOrien, float* afM0, float* afM1, float* afSymM){
/*
* calculate the misorientation betwen afM0 and afM1
* afMisOrien: iNM * iNSymM
* afM0: iNM * 9
* afM1: iNM * 9
* afSymM: symmetry matrix, iNSymM * 9
* NSymM: number of symmetry matrix
* call method: <<<(iNM,1),(iNSymM,1,1)>>>
*/
int i = blockIdx.x*blockDim.x + threadIdx.x;
float afTmp0[9];
float afTmp1[9];
float afM1Transpose[9];
float fCosAngle;
mat3_transpose(afM1Transpose, afM1 + blockIdx.x * 9);
mat3_dot(afTmp0, afSymM + threadIdx.x * 9, afM1Transpose);
mat3_dot(afTmp1, afM0 + blockIdx.x * 9, afTmp0);
fCosAngle = 0.5 * (afTmp1[0] + afTmp1[4] + afTmp1[8] - 1);
fCosAngle = min(0.9999999999, fCosAngle);
fCosAngle = max(-0.99999999999, fCosAngle);
afMisOrien[i] = acosf(fCosAngle);
}