#include "hip/hip_runtime.h"
#include "includes.h"
// tatami.cu


const unsigned nMax(100000000);
const unsigned nMaxSqrt(sqrt(nMax));



__global__ void odd(unsigned* v, unsigned base)
{
unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 7;
unsigned k2 = i + 3;
unsigned k3 = i + i - 4;
while ((k2 <= k3) && ((i * k2) < nMax))
{
unsigned k4 = (nMax - 1) / i;
if (k3 < k4)
k4 = k3;
__syncthreads();
for (unsigned j = k2 / 2; j <= k4 / 2; j++)
atomicInc(&v[i * j], 0xffffffff);
__syncthreads();
k2 += i + 1;
k3 += i - 1;
}
}