#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void Crossover(int *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__device__ void Crossover(char *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__device__ void Crossover(float *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__global__ void gpuCrossover(int *chromosome,hiprandState *globalState,int sizeofChromosome,int sizeofPopulation,int Bias,float prob){
int idx = blockIdx.x*blockDim.x+threadIdx.x;
int mid =(int) (generateRandom(globalState)*sizeofChromosome);//4;// (int) (generateRandom(globalState)*(sizeofChromosome-1));
//printf("MID: %d\n", mid);
idx=idx*2;
int start1,end1;

int start2,end2;
start1 = idx*sizeofChromosome;
end1 = start1+sizeofChromosome;
start2 = end1;
end2 = start2+sizeofChromosome;
if(end2<(sizeofChromosome*sizeofPopulation) )
Crossover(chromosome,sizeofChromosome,start1,end1,start2,end2,mid);
int number = (int) (generateRandom(globalState)*100);
if(number<(prob*100)){
int j = (int) (generateRandom(globalState)*((int)sizeofChromosome/4));
for(int k=0;k<j;k++){
int index = (int) (generateRandom(globalState)*sizeofChromosome);
int a = chromosome[index];// = //(chromosome[index]+1)%2;
if(a==1){
chromosome[index]=0;

}
else{
chromosome[index]=1;
}
}
}

}