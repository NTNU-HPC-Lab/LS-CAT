#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;


float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;




__device__ void Crossover(int *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__device__ void Crossover(char *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__device__ void Crossover(float *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

for(int i=mid;i<size;i++){
int c1 = start1+mid;
int c2 = start2+mid;
int temp = chromosome[c1];//
//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);

chromosome[c1]=	chromosome[c2];
chromosome[c2]=temp;
}
}
__device__ float generateRandomf( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__global__ void gpuCrossover(float *chromosome,hiprandState *globalState,int sizeofChromosome,int sizeofPopulation,int Bias,float prob){
int idx = blockIdx.x*blockDim.x+threadIdx.x;
int mid =(int) (generateRandomf(globalState)*sizeofChromosome);//4;// (int) (generateRandom(globalState)*(sizeofChromosome-1));
//printf("MID: %d\n", mid);
idx=idx*2;
int start1,end1;

int start2,end2;
start1 = idx*sizeofChromosome;
end1 = start1+sizeofChromosome;
start2 = end1;
end2 = start2+sizeofChromosome;
if(end2<(sizeofChromosome*sizeofPopulation) )
Crossover(chromosome,sizeofChromosome,start1,end1,start2,end2,mid);
int number = (int) (generateRandomf(globalState)*100);
if(number<(prob*100)){
int j = (int) (generateRandomf(globalState)*((int)sizeofChromosome/4));
for(int k=0;k<j;k++){
int index = (int) (generateRandomf(globalState)*sizeofChromosome);
float a = chromosome[index];// = //(chromosome[index]+1)%2;
if(a==1){
chromosome[index]=0;

}
else{
chromosome[index]=1;
}
}
}

}