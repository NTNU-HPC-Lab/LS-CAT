#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;


float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;




__global__ void setup_kernelf ( hiprandState *state, unsigned long seed )
{
hiprand_init ( seed, 0, 0, &state[0] );
}