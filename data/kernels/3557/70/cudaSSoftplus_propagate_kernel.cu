#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSoftplus_propagate_kernel(float* x, float* y, unsigned int size)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = log(1.0f + exp(x[i]));
}
}