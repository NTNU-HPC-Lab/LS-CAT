#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDTargetBiasPropagate_kernel( unsigned int size, const double bias, const double* inputs, const double* diffInputs, double* outputs)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
outputs[i] = inputs[i];

if (diffInputs[i] > 0.0 && inputs[i] > -bias)
outputs[i] += bias;
}
}