#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_weighted_kernel(unsigned int batchSize, unsigned int nbOutputs, unsigned int outputsHeight, unsigned int outputsWidth, float* estimated_labels, unsigned int nbChannels, unsigned int image_height, unsigned int image_width, float* input_image, unsigned char* workspace, float alpha)
{
const int batchEstimatedOffset = nbOutputs * outputsHeight * outputsWidth * blockIdx.z;
const int batchImageOffset = nbChannels * image_height * image_width * blockIdx.z;

const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < outputsWidth * outputsHeight; i += stride)
{
unsigned int outputMax = 0;

if (nbOutputs > 1)
{
float maxVal = estimated_labels[i + batchEstimatedOffset];

for (unsigned int cls = 1; cls < nbOutputs; ++cls) {
const float tmp = estimated_labels[i
+ cls*outputsWidth*outputsHeight
+ batchEstimatedOffset];

if (tmp > maxVal) {
outputMax = cls;
maxVal = tmp;
}
}
const unsigned char ch0
= (unsigned char) max(colors[outputMax%4][0]*alpha, min(255.0, colors[outputMax%4][0]*alpha + input_image[i + batchImageOffset]));
const unsigned char ch1
= (unsigned char) max(colors[outputMax%4][1]*alpha, min(255.0, colors[outputMax%4][1]*alpha + input_image[i + image_height*image_width + batchImageOffset]));
const unsigned char ch2
= (unsigned char) max(colors[outputMax%4][2]*alpha, min(255.0, colors[outputMax%4][2]*alpha + input_image[i + 2*image_height*image_width + batchImageOffset]));

workspace[i*3 + batchImageOffset] = ch0;
workspace[i*3 + 1 + batchImageOffset] = ch1;
workspace[i*3 + 2 + batchImageOffset] = ch2;
}
}
}