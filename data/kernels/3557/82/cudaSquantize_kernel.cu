#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSquantize_kernel(float* x, float* y, unsigned int size, float minVal, float maxVal, unsigned int quantizationLevels, bool truncate)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (quantizationLevels > 1) {
const float scaling = (maxVal - minVal)
/ (float)(quantizationLevels - 1);

for (unsigned int i = index; i < size; i += stride) {
const float clamped = (x[i] < minVal) ? minVal :
(x[i] > maxVal) ? maxVal :
x[i];

if (truncate)
y[i] = (int)((clamped - minVal) / scaling) * scaling + minVal;
else {
y[i] = (int)round((clamped - minVal) / scaling)
* scaling + minVal;
}
}
}
else {
for (unsigned int i = index; i < size; i += stride)
y[i] = ((x[i] >= 0.0f) ? 1.0f : -1.0f);
}
}