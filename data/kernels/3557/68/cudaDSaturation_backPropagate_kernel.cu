#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDSaturation_backPropagate_kernel(double* x, double* dx, unsigned int size, double threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (threshold != 0.0) {
dx[i] *= (x[i] > -threshold && x[i] < threshold)
? 1.0 : 0.0;
}
}
}