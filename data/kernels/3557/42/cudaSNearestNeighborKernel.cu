#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSNearestNeighborKernel(const float* input, size_t inputSizeX, size_t inputSizeY, float* output, size_t outputSizeX, size_t outputSizeY, size_t nbChannels, size_t batchSize)
{
const size_t inputOffset = (blockIdx.z*blockDim.z + threadIdx.z) * (nbChannels*inputSizeY*inputSizeX);
const size_t outputOffset = (blockIdx.z*blockDim.z + threadIdx.z) * (nbChannels*outputSizeY*outputSizeX);

const float multy = ((float) inputSizeY)/((float) outputSizeY);
const float multx = ((float) inputSizeX)/((float) outputSizeX);

for(size_t channel = blockIdx.x; channel < nbChannels; channel += gridDim.x) {
for(size_t oy = threadIdx.y; oy < outputSizeY; oy += blockDim.y) {
for(size_t ox = threadIdx.x; ox < outputSizeX; ox += blockDim.x) {
const size_t iy = (size_t) oy*multy;
const size_t ix = (size_t) ox*multx;


output[outputOffset +
channel*outputSizeY*outputSizeX +
oy*outputSizeX +
ox] = input[inputOffset +
channel*inputSizeY*inputSizeX +
iy*inputSizeX +
ix];

}
}
}
}