#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUpdateMetrics_kernel(float * inputs, int * activity, long long unsigned int * firingRate, long long unsigned int * totalFiringRate, long long int * outputsActivity, long long int * totalOutputsActivity, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
const unsigned int batchInputOffset = blockIdx.z * inputSize;

for (unsigned int channel = blockIdx.x; channel < inputsDimZ; channel += gridDim.x) {
for (unsigned int y = threadIdx.y; y < inputsDimY; y += blockDim.y) {
for (unsigned int x = threadIdx.x; x < inputsDimX; x += blockDim.x) {

const unsigned int inputsIdx =
x + y*inputsDimX + channel*inputsDimX*inputsDimY;

int value = round(inputs[inputsIdx + batchInputOffset]);
unsigned int event = value == 0 ? 0 : 1;

activity[inputsIdx + batchInputOffset] = event;
firingRate[inputsIdx + batchInputOffset] += event;
totalFiringRate[inputsIdx + batchInputOffset] += event;
outputsActivity[inputsIdx + batchInputOffset] += value;
totalOutputsActivity[inputsIdx + batchInputOffset] += value;
}
}
}
}