#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSMaxForward_kernel(unsigned int size, float* input, float* maxVal, const unsigned int idx, unsigned int* argMax)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (input[i] > maxVal[i]) {
maxVal[i] = input[i];
argMax[i] = idx;
}
}
}