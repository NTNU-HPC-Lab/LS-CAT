#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSqrt_kernel(unsigned int size, float* data)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride)
data[i] = sqrt(data[i]);
}