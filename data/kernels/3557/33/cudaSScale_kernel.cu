#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSScale_kernel(unsigned int size, float* input, const float scale, const float shift, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride)
result[i] = input[i] * scale + shift + beta * result[i];
}
else {
for (unsigned int i = index; i < size; i += stride)
result[i] = input[i] * scale  + shift;
}
}