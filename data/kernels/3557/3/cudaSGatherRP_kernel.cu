#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSGatherRP_kernel(   unsigned int inputSizeX, unsigned int inputSizeY, unsigned int nbAnchors, unsigned int batchSize, const float* inputs, const float* i, const float* j, const float* k, const float* b, const int* mask, float* outputs, const unsigned int topN, const unsigned int nbProposals)
{
const int batchPos = blockIdx.z;
const int sortOffset = batchPos*topN;

int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

const int totalIndex = index + sortOffset;
const int batchIndex = index + batchPos*nbProposals;

if(index < nbProposals)
{
unsigned int xIdx = i[ mask[totalIndex] + sortOffset ]
+ j[mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[mask[totalIndex] + sortOffset ] + nbAnchors)*inputSizeX*inputSizeY
+ b[mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int yIdx = i[mask[totalIndex] + sortOffset ]
+ j[mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[mask[totalIndex] + sortOffset ] + 2*nbAnchors)*inputSizeX*inputSizeY
+ b[mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int wIdx = i[mask[totalIndex] + sortOffset ]
+ j[mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[mask[totalIndex] + sortOffset ] + 3*nbAnchors)*inputSizeX*inputSizeY
+ b[mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int hIdx = i[mask[totalIndex] + sortOffset ]
+ j[mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[mask[totalIndex] + sortOffset ] + 4*nbAnchors)*inputSizeX*inputSizeY
+ b[mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

outputs[0 + (batchIndex)*4] = inputs[xIdx];
outputs[1 + (batchIndex)*4] = inputs[yIdx];
outputs[2 + (batchIndex)*4] = inputs[wIdx];
outputs[3 + (batchIndex)*4] = inputs[hIdx];
}

}