#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void spatial_output_kernel(unsigned int nbClass, unsigned int targetHeight, unsigned int targetWidth, float threshold, float* targetData, uint32_t* outputEstimated)
{
const int batchInputOffset = targetWidth * targetHeight * nbClass * blockIdx.z;
const int batchOutputOffset = targetWidth * targetHeight * blockIdx.z;

const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < targetWidth * targetHeight; i += stride)
{
unsigned int outputMax = 0;

if (nbClass > 1)
{
float maxVal = targetData[i + batchInputOffset];

for (unsigned int cls = 1; cls < nbClass; ++cls) {
const float tmp = targetData[i + cls*targetWidth*targetHeight
+ batchInputOffset];

if (tmp > maxVal) {
outputMax = cls;
maxVal = tmp;
}
}

outputEstimated[i + batchOutputOffset] = outputMax;
}
else if(nbClass == 1)
{
if(targetData[index] > threshold)
outputMax = 1;

const int estimatedLabel
= (targetData[i + batchInputOffset] > threshold);

outputEstimated[i + batchOutputOffset] = estimatedLabel;

}
}
}