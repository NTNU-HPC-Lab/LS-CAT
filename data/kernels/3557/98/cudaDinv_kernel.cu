#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDinv_kernel(unsigned int size, const double *x, double *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = 1.0 / x[i];
}
}