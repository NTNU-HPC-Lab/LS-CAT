#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cudaSNormalizeROIs_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned int inputSizeX = 1;
unsigned int inputSizeY = 1;
unsigned int nbProposals = 1;
unsigned int batchSize = 1;
unsigned int scoreIdx = 1;
unsigned int nbCls = 1;
unsigned int maxParts = 1;
unsigned int maxTemplates = 1;
bool keepMax = 1;
bool generateParts = 1;
bool generateTemplates = 1;
const float normX = 1;
const float normY = 1;
const float *means = NULL;
hipMalloc(&means, XSIZE*YSIZE);
const float *std = NULL;
hipMalloc(&std, XSIZE*YSIZE);
const unsigned int *numPartsPerClass = NULL;
hipMalloc(&numPartsPerClass, XSIZE*YSIZE);
const unsigned int *numTemplatesPerClass = NULL;
hipMalloc(&numTemplatesPerClass, XSIZE*YSIZE);
const float *ROIRef = NULL;
hipMalloc(&ROIRef, XSIZE*YSIZE);
const float *ROIEst = NULL;
hipMalloc(&ROIEst, XSIZE*YSIZE);
const float *ValuesEst = NULL;
hipMalloc(&ValuesEst, XSIZE*YSIZE);
const float *partsEst = NULL;
hipMalloc(&partsEst, XSIZE*YSIZE);
const float *partsVisibilityEst = NULL;
hipMalloc(&partsVisibilityEst, XSIZE*YSIZE);
const float *templatesEst = NULL;
hipMalloc(&templatesEst, XSIZE*YSIZE);
float *outputs = NULL;
hipMalloc(&outputs, XSIZE*YSIZE);
int *argMax = NULL;
hipMalloc(&argMax, XSIZE*YSIZE);
float *partsPrediction = NULL;
hipMalloc(&partsPrediction, XSIZE*YSIZE);
float *partsVisibilityPrediction = NULL;
hipMalloc(&partsVisibilityPrediction, XSIZE*YSIZE);
float *templatesPrediction = NULL;
hipMalloc(&templatesPrediction, XSIZE*YSIZE);
float scoreThreshold = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cudaSNormalizeROIs_kernel<<<gridBlock,threadBlock>>>(inputSizeX,inputSizeY,nbProposals,batchSize,scoreIdx,nbCls,maxParts,maxTemplates,keepMax,generateParts,generateTemplates,normX,normY,means,std,numPartsPerClass,numTemplatesPerClass,ROIRef,ROIEst,ValuesEst,partsEst,partsVisibilityEst,templatesEst,outputs,argMax,partsPrediction,partsVisibilityPrediction,templatesPrediction,scoreThreshold);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cudaSNormalizeROIs_kernel<<<gridBlock,threadBlock>>>(inputSizeX,inputSizeY,nbProposals,batchSize,scoreIdx,nbCls,maxParts,maxTemplates,keepMax,generateParts,generateTemplates,normX,normY,means,std,numPartsPerClass,numTemplatesPerClass,ROIRef,ROIEst,ValuesEst,partsEst,partsVisibilityEst,templatesEst,outputs,argMax,partsPrediction,partsVisibilityPrediction,templatesPrediction,scoreThreshold);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cudaSNormalizeROIs_kernel<<<gridBlock,threadBlock>>>(inputSizeX,inputSizeY,nbProposals,batchSize,scoreIdx,nbCls,maxParts,maxTemplates,keepMax,generateParts,generateTemplates,normX,normY,means,std,numPartsPerClass,numTemplatesPerClass,ROIRef,ROIEst,ValuesEst,partsEst,partsVisibilityEst,templatesEst,outputs,argMax,partsPrediction,partsVisibilityPrediction,templatesPrediction,scoreThreshold);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}