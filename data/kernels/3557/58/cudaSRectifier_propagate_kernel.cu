#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSRectifier_propagate_kernel(float* x, float* y, unsigned int size, float leakSlope, float clipping)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
float value = x[i];

if (clipping > 0.0f)
y[i] = (value > 0.0f) ? min(value, clipping) : leakSlope * value;
else
y[i] = (value > 0.0f) ? value : leakSlope * value;
}
}