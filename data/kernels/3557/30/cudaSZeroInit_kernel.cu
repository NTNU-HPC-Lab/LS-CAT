#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSZeroInit_kernel(unsigned int size, float* data)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride)
data[i] = 0.0f;
}