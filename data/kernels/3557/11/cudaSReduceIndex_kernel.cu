#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSReduceIndex_kernel(  const unsigned int inputSize, const unsigned int inputBatchOffset, const unsigned int outputBatchOffset, const unsigned int channelsWidth, const unsigned int channelsHeight, const unsigned int nbAnchors, const float* valueThreshold, const float* inputs, int* outputMap, float* scores)
{
const int batchPos = blockIdx.z;
const int clsPos = blockIdx.y;

const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

const int inputIndex = index
+ inputSize*blockIdx.y
+ batchPos*inputBatchOffset;

const int outputIndex = index
+ inputSize*blockIdx.y
+ batchPos*outputBatchOffset;

if(index < inputSize)
{
float value = inputs[inputIndex];

if(value >= valueThreshold[clsPos])
{
outputMap[outputIndex] = index;
scores[outputIndex] = value;
}
else
{
outputMap[outputIndex] = -255;
scores[outputIndex] = -255.0;
}
}
}