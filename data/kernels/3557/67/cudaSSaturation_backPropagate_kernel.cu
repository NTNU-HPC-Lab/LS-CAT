#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSaturation_backPropagate_kernel(float* x, float* dx, unsigned int size, float threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (threshold != 0.0f) {
dx[i] *= (x[i] > -threshold && x[i] < threshold)
? 1.0f : 0.0f;
}
}
}