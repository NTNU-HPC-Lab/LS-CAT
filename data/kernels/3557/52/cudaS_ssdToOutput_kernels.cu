#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaS_ssdToOutput_kernels(  unsigned int batchSize, unsigned int nbClass, unsigned int nbAnchors, unsigned int channelWidth, unsigned int channelHeight, unsigned int nbProposals, unsigned int* nbValidROIs, unsigned int cls, unsigned int totalParts, unsigned int totalTemplates, unsigned int maxParts, unsigned int maxTemplates, unsigned int cumulParts, unsigned int cumulTemplates, unsigned int nbParts, unsigned int nbTemplates, float xRatio, float yRatio, float xOutputRatio, float yOutputRatio, const float* roi_bbox, const float* roi_anchors, const float* anchors, const float* inputs_parts, const float* inputs_templates, float* outputs)
{
const int batchPos = blockIdx.z;
const int proposal = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;
const int ptIdx = blockIdx.y;
const unsigned int nbAnchorPerCls = nbAnchors;

const int nbDetectedObject  = (int) nbValidROIs[batchPos];
const int nbIdx = 6;
if(proposal < nbProposals)
{
const unsigned int n = proposal + cls*nbProposals + batchPos*nbProposals*nbClass;

if(proposal < nbDetectedObject)
{
if(ptIdx == 0)
{
outputs[0 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = roi_bbox[0 + 5*proposal + batchPos*nbProposals*5];
outputs[1 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = roi_bbox[1 + 5*proposal + batchPos*nbProposals*5];
outputs[2 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = roi_bbox[2 + 5*proposal + batchPos*nbProposals*5];
outputs[3 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = roi_bbox[3 + 5*proposal + batchPos*nbProposals*5];
outputs[4 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = roi_bbox[4 + 5*proposal + batchPos*nbProposals*5];
outputs[5 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = (float) cls;
}

if(ptIdx < nbParts && totalParts > 0)
{
const unsigned int xa   = roi_anchors[0 + 5*proposal + batchPos*nbProposals*5];
const unsigned int ya   = roi_anchors[1 + 5*proposal + batchPos*nbProposals*5];
const unsigned int k    = roi_anchors[2 + 5*proposal + batchPos*nbProposals*5];


const int yIdx = xa
+ ya*channelWidth
+ (k*nbParts*2 + cumulParts + ptIdx*2)*channelHeight*channelWidth
+ batchPos*channelHeight*channelWidth*nbAnchorPerCls*2*totalParts;
const int xIdx = xa
+ ya*channelWidth
+ (k*nbParts*2 + cumulParts + ptIdx*2 + 1)*channelHeight*channelWidth
+ batchPos*channelHeight*channelWidth*nbAnchorPerCls*2*totalParts;


const float partY = inputs_parts[yIdx];
const float partX = inputs_parts[xIdx];

const int xa0 = (int)(anchors[cls*4*nbAnchorPerCls + k*4] + xa * xRatio);
const int ya0 = (int)(anchors[cls*4*nbAnchorPerCls + k*4 + 1] + ya * yRatio);
const int xa1 = (int)(anchors[cls*4*nbAnchorPerCls + k*4 + 2] + xa * xRatio);
const int ya1 = (int)(anchors[cls*4*nbAnchorPerCls + k*4 + 3] + ya * yRatio);

// Anchors width and height
const int wa = xa1 - xa0;
const int ha = ya1 - ya0;

// Anchor center coordinates (xac, yac)
const float xac = xa0 + wa / 2.0;
const float yac = ya0 + ha / 2.0;
const float predPartY = ((partY) * ha + yac)*yOutputRatio ;
const float predPartX = ((partX) * wa + xac)*xOutputRatio ;

outputs[ptIdx*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = predPartY;
outputs[ptIdx*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = predPartX;

}
else if(ptIdx < maxParts && totalParts > 0)
{
outputs[ptIdx*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
}

///for(unsigned int t = 0; t < nbTemplates; ++t)
if(ptIdx < nbTemplates && totalTemplates > 0)
{
const unsigned int xa   = roi_anchors[0 + 5*proposal + batchPos*nbProposals*5];
const unsigned int ya   = roi_anchors[1 + 5*proposal + batchPos*nbProposals*5];
const unsigned int k    = roi_anchors[2 + 5*proposal + batchPos*nbProposals*5];

const int yIdx = xa
+ ya*channelWidth
+ (k*nbTemplates*3 + cumulTemplates + ptIdx*3)*channelHeight*channelWidth
+ batchPos*channelHeight*channelWidth*nbAnchorPerCls*3*totalTemplates;
const int xIdx = xa
+ ya*channelWidth
+ (k*nbTemplates*3 + cumulTemplates + ptIdx*3 + 1)*channelHeight*channelWidth
+ batchPos*channelHeight*channelWidth*nbAnchorPerCls*3*totalTemplates;
const int zIdx = xa
+ ya*channelWidth
+ (k*nbTemplates*3 + cumulTemplates + ptIdx*3 + 2)*channelHeight*channelWidth
+ batchPos*channelHeight*channelWidth*nbAnchorPerCls*3*totalTemplates;


const float templateY = expf(inputs_templates[yIdx]);
const float templateX = expf(inputs_templates[xIdx]);
const float templateZ = expf(inputs_templates[zIdx]);

outputs[ptIdx*3 + maxParts*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = templateY;
outputs[ptIdx*3 + maxParts*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = templateX;
outputs[ptIdx*3 + maxParts*2 + 2 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = templateZ;

}
else if(ptIdx < maxTemplates && totalTemplates > 0)
{
outputs[ptIdx*3 + maxParts*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*3 + maxParts*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*3 + maxParts*2 + 2 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
}

}
else
{
outputs[0 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = 0.0;
outputs[1 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = 0.0;
outputs[2 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = 0.0;
outputs[3 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = 0.0;
outputs[4 + n*(nbIdx + maxParts*2 + maxTemplates*3)] = 0.0;

//for(unsigned int p = 0; p < nbParts; ++p)
if(ptIdx < maxParts && totalParts > 0)
{
outputs[ptIdx*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
}

//for(unsigned int t = 0;t < nbTemplates; ++t)
if(ptIdx < maxTemplates && totalTemplates > 0)
{
outputs[ptIdx*3 + maxParts*2 + 0 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*3 + maxParts*2 + 1 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
outputs[ptIdx*3 + maxParts*2 + 2 + nbIdx + n*(nbIdx + maxParts*2 + maxTemplates*3) ] = 0.0;
}

}
}
}