#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDadd_kernel(unsigned int size, double value, const double *x, double *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = x[i] + value;
}
}