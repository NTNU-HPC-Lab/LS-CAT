#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSaturation_propagate_kernel(float* x, float* y, unsigned int size, float threshold)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
float value = x[i];

if (threshold != 0.0f) {
y[i] = (value < -threshold) ? -threshold
: (value > threshold) ? threshold
: value;
}
}
}