#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDpow_kernel(unsigned int size, double power, const double *x, double *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = powf(x[i], power);
}
}