#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSRectifier_backPropagate_kernel(float* x, float* dx, unsigned int size, float leakSlope, float clipping)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
if (clipping > 0.0) {
dx[i] *= (x[i] > clipping) ? 0.0f : (x[i] > 0.0f)
? 1.0f
: leakSlope;
}
else
dx[i] *= (x[i] > 0.0f) ? 1.0f : leakSlope;
}
}