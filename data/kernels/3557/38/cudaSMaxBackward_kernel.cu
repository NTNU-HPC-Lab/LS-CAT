#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSMaxBackward_kernel(unsigned int size, float* diffInput, const unsigned int idx, unsigned int* argMax, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride) {
result[i] = (argMax[i] == idx) ? (diffInput[i] + beta * result[i])
: beta * result[i];
}
}
else {
for (unsigned int i = index; i < size; i += stride) {
result[i] = (argMax[i] == idx) ? diffInput[i]
: 0.0f;
}
}
}