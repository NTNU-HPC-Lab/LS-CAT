#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSGatherRP_kernel( unsigned int inputSizeX, unsigned int inputSizeY, unsigned int nbAnchors, unsigned int batchSize, const float* inputs, const int* i, const int* j, const int* k, const int* b, const int* mask, float* outputs, int* anchors, unsigned int topN, const unsigned int nbProposals)
{
const int batchPos = blockIdx.z;
const int sortOffset = batchPos*topN;
const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;
const int totalIndex = index + sortOffset;
const int batchIndex = index + batchPos*nbProposals;

if(index < nbProposals)
{
unsigned int xIdx = i[ mask[totalIndex] + sortOffset ]
+ j[ mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[ mask[totalIndex] + sortOffset ] + nbAnchors)*inputSizeX*inputSizeY
+ b[ mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int yIdx = i[ mask[totalIndex] + sortOffset ]
+ j[ mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[ mask[totalIndex] + sortOffset ] + 2*nbAnchors)*inputSizeX*inputSizeY
+ b[ mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int wIdx = i[ mask[totalIndex] + sortOffset ]
+ j[ mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[ mask[totalIndex] + sortOffset ] + 3*nbAnchors)*inputSizeX*inputSizeY
+ b[ mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

unsigned int hIdx = i[ mask[totalIndex] + sortOffset ]
+ j[ mask[totalIndex] + sortOffset ]*inputSizeX
+ (k[ mask[totalIndex] + sortOffset ] + 4*nbAnchors)*inputSizeX*inputSizeY
+ b[ mask[totalIndex] + sortOffset ]*nbAnchors*inputSizeX*inputSizeY*6;

anchors[0 + (batchIndex)*4] = i[mask[totalIndex]];
anchors[1 + (batchIndex)*4] = j[mask[totalIndex]];
anchors[2 + (batchIndex)*4] = k[mask[totalIndex]];
anchors[3 + (batchIndex)*4] = b[mask[totalIndex]];

outputs[0 + (batchIndex)*4] = inputs[xIdx];
outputs[1 + (batchIndex)*4] = inputs[yIdx];
outputs[2 + (batchIndex)*4] = inputs[wIdx];
outputs[3 + (batchIndex)*4] = inputs[hIdx];
}

}