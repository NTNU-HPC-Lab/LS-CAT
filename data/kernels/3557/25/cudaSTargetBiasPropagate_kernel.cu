#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSTargetBiasPropagate_kernel( unsigned int size, const float bias, const float* inputs, const float* diffInputs, float* outputs)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
outputs[i] = inputs[i];

if (diffInputs[i] > 0.0f && inputs[i] > -bias)
outputs[i] += bias;
}
}