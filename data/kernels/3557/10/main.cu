#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cudaS_ssdToOutput_kernels.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned int batchSize = 1;
unsigned int nbClass = 1;
unsigned int nbAnchors = 1;
unsigned int channelWidth = 1;
unsigned int channelHeight = 1;
unsigned int nbProposals = 1;
unsigned int *nbValidROIs = NULL;
hipMalloc(&nbValidROIs, XSIZE*YSIZE);
unsigned int cls = 1;
unsigned int totalParts = 1;
unsigned int totalTemplates = 1;
unsigned int maxParts = 1;
unsigned int maxTemplates = 1;
unsigned int cumulParts = 1;
unsigned int cumulTemplates = 1;
unsigned int nbParts = 1;
unsigned int nbTemplates = 1;
float xRatio = 1;
float yRatio = 1;
float xOutputRatio = 1;
float yOutputRatio = 1;
const float *roi_bbox = NULL;
hipMalloc(&roi_bbox, XSIZE*YSIZE);
const float *roi_anchors = NULL;
hipMalloc(&roi_anchors, XSIZE*YSIZE);
const float *anchors = NULL;
hipMalloc(&anchors, XSIZE*YSIZE);
const float *inputs_parts = NULL;
hipMalloc(&inputs_parts, XSIZE*YSIZE);
const float *inputs_templates = NULL;
hipMalloc(&inputs_templates, XSIZE*YSIZE);
float *outputs = NULL;
hipMalloc(&outputs, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cudaS_ssdToOutput_kernels<<<gridBlock,threadBlock>>>(batchSize,nbClass,nbAnchors,channelWidth,channelHeight,nbProposals,nbValidROIs,cls,totalParts,totalTemplates,maxParts,maxTemplates,cumulParts,cumulTemplates,nbParts,nbTemplates,xRatio,yRatio,xOutputRatio,yOutputRatio,roi_bbox,roi_anchors,anchors,inputs_parts,inputs_templates,outputs);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cudaS_ssdToOutput_kernels<<<gridBlock,threadBlock>>>(batchSize,nbClass,nbAnchors,channelWidth,channelHeight,nbProposals,nbValidROIs,cls,totalParts,totalTemplates,maxParts,maxTemplates,cumulParts,cumulTemplates,nbParts,nbTemplates,xRatio,yRatio,xOutputRatio,yOutputRatio,roi_bbox,roi_anchors,anchors,inputs_parts,inputs_templates,outputs);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cudaS_ssdToOutput_kernels<<<gridBlock,threadBlock>>>(batchSize,nbClass,nbAnchors,channelWidth,channelHeight,nbProposals,nbValidROIs,cls,totalParts,totalTemplates,maxParts,maxTemplates,cumulParts,cumulTemplates,nbParts,nbTemplates,xRatio,yRatio,xOutputRatio,yOutputRatio,roi_bbox,roi_anchors,anchors,inputs_parts,inputs_templates,outputs);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}