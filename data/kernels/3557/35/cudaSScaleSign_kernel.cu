#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSScaleSign_kernel(unsigned int size, float* input, float* sign, const float scale, const float beta, float* result)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

if (beta != 0.0f) {
for (unsigned int i = index; i < size; i += stride) {
const float sgn = (sign[i] >= 0) ? 1.0f : -1.0f;
result[i] = input[i] * sgn * scale + beta * result[i];
}
}
else {
for (unsigned int i = index; i < size; i += stride) {
const float sgn = (sign[i] >= 0) ? 1.0f : -1.0f;
result[i] = input[i] * sgn * scale;
}
}
}