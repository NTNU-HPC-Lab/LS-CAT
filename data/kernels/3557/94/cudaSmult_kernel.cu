#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSmult_kernel(unsigned int size, const float *x1, const float *x2, float *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = x1[i] * x2[i];
}
}