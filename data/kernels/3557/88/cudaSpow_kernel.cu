#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSpow_kernel(unsigned int size, float power, const float *x, float *y)
{
const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = index; i < size; i += stride) {
y[i] = powf(x[i], power);
}
}