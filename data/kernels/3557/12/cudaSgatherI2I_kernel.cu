#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSgatherI2I_kernel( const int* keys, const int* indicesX, const int* indicesY, const int* indicesK, int* outX, int* outY, int* outK, unsigned int nbElements)
{
const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

if(index < nbElements)
{
const int key = keys[index];
printf("keys[%d]=%d indicesX[%d]:%d  ", index, key, index, indicesX[index] );
outX[index] = indicesX[key];
outY[index] = indicesY[key];
outK[index] = indicesK[key];
}
}