#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kUpdate(int nbSpx, float* clusters, float* accAtt_g)
{
int cluster_idx = blockIdx.x*blockDim.x + threadIdx.x;

if (cluster_idx<nbSpx){
int nbSpx2 = nbSpx * 2;
int nbSpx3 = nbSpx * 3;
int nbSpx4 = nbSpx * 4;
int nbSpx5 = nbSpx * 5;
int counter = accAtt_g[cluster_idx + nbSpx5];
if (counter != 0){
clusters[cluster_idx] = accAtt_g[cluster_idx] / counter;
clusters[cluster_idx + nbSpx] = accAtt_g[cluster_idx + nbSpx] / counter;
clusters[cluster_idx + nbSpx2] = accAtt_g[cluster_idx + nbSpx2] / counter;
clusters[cluster_idx + nbSpx3] = accAtt_g[cluster_idx + nbSpx3] / counter;
clusters[cluster_idx + nbSpx4] = accAtt_g[cluster_idx + nbSpx4] / counter;

//reset accumulator
accAtt_g[cluster_idx] = 0;
accAtt_g[cluster_idx + nbSpx] = 0;
accAtt_g[cluster_idx + nbSpx2] = 0;
accAtt_g[cluster_idx + nbSpx3] = 0;
accAtt_g[cluster_idx + nbSpx4] = 0;
accAtt_g[cluster_idx + nbSpx5] = 0;
}
}
}