#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
for (int i=blockIdx.x;i<b;i+=gridDim.x){
for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
float x1=xyz1[(i*n+j)*5+0];
float y1=xyz1[(i*n+j)*5+1];
float r1=xyz1[(i*n+j)*5+2];
float g1=xyz1[(i*n+j)*5+3];
float b1=xyz1[(i*n+j)*5+4];
int j2=idx1[i*n+j];
float x2=xyz2[(i*m+j2)*5+0];
float y2=xyz2[(i*m+j2)*5+1];
float r2=xyz2[(i*m+j2)*5+2];
float g2=xyz2[(i*m+j2)*5+3];
float b2=xyz2[(i*m+j2)*5+4];
float g=grad_dist1[i*n+j]*2;
atomicAdd(&(grad_xyz1[(i*n+j)*5+0]),g*(x1-x2));
atomicAdd(&(grad_xyz1[(i*n+j)*5+1]),g*(y1-y2));
atomicAdd(&(grad_xyz1[(i*n+j)*5+2]),g*(r1-r2));
atomicAdd(&(grad_xyz1[(i*n+j)*5+3]),g*(g1-g2));
atomicAdd(&(grad_xyz1[(i*n+j)*5+4]),g*(b1-b2));
atomicAdd(&(grad_xyz2[(i*m+j2)*5+0]),-(g*(x1-x2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*5+1]),-(g*(y1-y2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*5+2]),-(g*(r1-r2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*5+3]),-(g*(g1-g2)));
atomicAdd(&(grad_xyz2[(i*m+j2)*5+4]),-(g*(b1-b2)));
}
}
}