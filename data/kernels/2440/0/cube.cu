#include "hip/hip_runtime.h"
#include "includes.h"

//this function is a kernel
//__global__ is a C construct called declaration specifier and that’s how CUDA knows that this is not CPU code but a kernel
//threadIdx: CUDA has a built in variable called threadIdx which tells each thread its index within a block. Its a C construct
//with 3 members “x”, “y” and “z” and the struct is called “dim3"

__global__ void cube(float * d_out, float * d_in){
int idx = threadIdx.x; //
float f = d_in[idx];
d_out[idx] = f*f*f;
}