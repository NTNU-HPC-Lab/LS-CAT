#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= n) return;
binary[i] = (x[i] > 0) ? 1 : -1;
}