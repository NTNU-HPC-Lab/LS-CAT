#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
{
int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (f >= n) return;
int i = 0;
float mean = 0;
for(i = 0; i < size; ++i){
mean += abs(filters[f*size + i]);
}
mean = mean / size;
for(i = 0; i < size; ++i){
binary[f*size + i] = (filters[f*size + i] > 0) ? mean : -mean;
}
}