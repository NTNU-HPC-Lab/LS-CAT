#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matchHistCuda(float*qSet, float*dbSet, size_t qSize, size_t dbSize, size_t hSize, float*out){
size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
size_t idy = blockIdx.y*blockDim.y + threadIdx.y;

if(idx < qSize && idy < dbSize){
size_t qi = idx*hSize;
size_t dbi = idy*hSize;

//Cosine similarity code ------------
float sumab = 0;
float suma2 = 0;
float sumb2 = 0;

for(int k = 0; k < hSize; k++){
sumab += qSet[qi+k] * dbSet[dbi+k];
suma2 += qSet[qi+k] * qSet[qi+k];
sumb2 += dbSet[dbi+k] * dbSet[dbi+k];
}

float cossim = sumab/(sqrtf(suma2)/sqrtf(sumb2));
out[idy*qSize + idx] = cossim;
}
}