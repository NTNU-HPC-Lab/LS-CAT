#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rowMin(float* input, int* output, size_t rowS, size_t rowNum){
size_t id = blockIdx.x*blockDim.x + threadIdx.x;

if(id < rowNum){
float temp[MAX_K/2][2];
size_t inId = id * rowS;

for(int i = 0; i< rowS;i++){
temp[i][0] = input[inId + i];
temp[i][1] = (float)i;
}

for(int i = 0; i< rowS; i++){
float best = temp[i][0];
int bestInd = i;
for(int j = i; j < rowS; j++){
if(temp[j][0] > best){
best = temp[j][0];
bestInd = j;
}
}
float iVal = temp[i][0];
float iInd = temp[i][1];
temp[i][0] = temp[bestInd][0];
temp[i][1] = temp[bestInd][1];
temp[bestInd][0] = iVal;
temp[bestInd][1] = iInd;
}

for(int i = 0; i< rowS; i++){
output[inId+i] = (int)temp[i][1];
}
}
}