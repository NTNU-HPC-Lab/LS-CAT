#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createHistCuda (float* siftCentroids, float* siftImage, int linesCent, int linesIm, float* temp)
{
__shared__ float cosines[BLOCK_SIZE][2];

size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
size_t idy = blockIdx.y;
size_t tid = threadIdx.x;

if(idx < linesCent){
int centin = idx * 128;
int imin = idy * 128;

//Cosine similarity code ------------
float sumab = 0;
float suma2 = 0;
float sumb2 = 0;

for(int k = 0; k < 128; k++){
sumab += siftCentroids[centin + k] * siftImage[imin + k];
suma2 += siftImage[imin + k] * siftImage[imin + k];
sumb2 += siftCentroids[centin + k] * siftCentroids[centin + k];
}

float cossim = sumab/(sqrtf(suma2)/sqrtf(sumb2));

//debug[idy*linesCent + idx] = cossim;
cosines[threadIdx.x][0] = cossim;
cosines[threadIdx.x][1] = idx;

__syncthreads();

for (unsigned int s=blockDim.x/2; s>0; s>>=1)
{
if (tid < s){
size_t tid2 = tid + s;
if(cosines[tid2][0] > cosines[tid][0]){
cosines[tid][0] = cosines[tid2][0];
cosines[tid][1] = cosines[tid2][1];
}
}
__syncthreads();
}

if (tid == 0){
temp[(blockIdx.y*gridDim.x + blockIdx.x)*2] = cosines[0][0];
temp[(blockIdx.y*gridDim.x + blockIdx.x)*2+1] = cosines[0][1];
}

}

}