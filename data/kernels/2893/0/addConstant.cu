#include "hip/hip_runtime.h"
#include "includes.h"



#define DEBUG false
#define DEBUG_OUTPUT false
#define DEBUG_DELTA_K false
#define DEBUGNET false
#define DEBUG_TIMEING true
#define index(i,j,ld) (((j)*(ld))+(i))

int numBlocks = 1;
int blockSize = 256;

using namespace std;

/*
*  Print Matrix on host
*/
__global__ void addConstant(float* input, float constant, int num_elements){
const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if(tid < num_elements)
{
input[tid] = input[tid] + constant;
}
}