#include "hip/hip_runtime.h"
#include "includes.h"



#define DEBUG false
#define DEBUG_OUTPUT false
#define DEBUG_DELTA_K false
#define DEBUGNET false
#define DEBUG_TIMEING true
#define index(i,j,ld) (((j)*(ld))+(i))

int numBlocks = 1;
int blockSize = 256;

using namespace std;

/*
*  Print Matrix on host
*/
__global__ void squaredError(float* predicted_values, float* actual_values, float* results, int num_elements){
const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if(tid < num_elements){
float value = pow(actual_values[tid] - predicted_values[tid], 2.0);
results[tid] = value;
}
}