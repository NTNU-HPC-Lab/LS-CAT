#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void down_sweep_512( uint* data_block ) {
for (uint i=512; i>=2; i>>=1) {
for (uint j=0; j<(511 + blockDim.x) / i; ++j) {
const auto element = 511 - (j*blockDim.x + threadIdx.x) * i;
if (element < 512) {
const auto other_element = element - (i>>1);
const auto value = data_block[other_element];
data_block[other_element] = data_block[element];
data_block[element] += value;
}
}
__syncthreads();
}
}
__device__ void up_sweep_512( uint* data_block ) {
uint starting_elem = 1;
for (uint i=2; i<=512; i<<=1) {
for (uint j=0; j<(511 + blockDim.x) / i; ++j) {
const uint element = starting_elem + (j*blockDim.x + threadIdx.x) * i;
if (element < 512) {
data_block[element] += data_block[element - (i>>1)];
}
}
starting_elem += i;
__syncthreads();
}
}
__global__ void prefix_sum_reduce( uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size ) {
// Use a data block size of 512
__shared__ uint data_block [512];

// Let's do it in blocks of 512 (2^9)
const uint last_block = array_size >> 9;
if (blockIdx.x < last_block) {
const uint first_elem = blockIdx.x << 9;

// Load elements into shared memory, add prev_last_elem
data_block[threadIdx.x] = dev_main_array[first_elem + threadIdx.x];
data_block[threadIdx.x + blockDim.x] = dev_main_array[first_elem + threadIdx.x + blockDim.x];

__syncthreads();

up_sweep_512((uint*) &data_block[0]);

if (threadIdx.x == 0) {
dev_auxiliary_array[blockIdx.x] = data_block[511];
data_block[511] = 0;
}

__syncthreads();

down_sweep_512((uint*) &data_block[0]);

// Store back elements
//assert( first_elem + threadIdx.x + blockDim.x < number_of_events * VeloTracking::n_modules + 2);
dev_main_array[first_elem + threadIdx.x] = data_block[threadIdx.x];
dev_main_array[first_elem + threadIdx.x + blockDim.x] = data_block[threadIdx.x + blockDim.x];

__syncthreads();
}

// Last block is special because
// it may contain an unspecified number of elements
else {
const auto elements_remaining = array_size & 0x1FF; // % 512
if (elements_remaining > 0) {
const auto first_elem = array_size - elements_remaining;

// Initialize all elements to zero
data_block[threadIdx.x] = 0;
data_block[threadIdx.x + blockDim.x] = 0;

// Load elements
const auto elem_index = first_elem + threadIdx.x;
if (elem_index < array_size) {
data_block[threadIdx.x] = dev_main_array[elem_index];
}
if ((elem_index+blockDim.x) < array_size) {
data_block[threadIdx.x + blockDim.x] = dev_main_array[elem_index + blockDim.x];
}

__syncthreads();

up_sweep_512((uint*) &data_block[0]);

// Store sum of all elements
if (threadIdx.x == 0) {
dev_auxiliary_array[blockIdx.x] = data_block[511];
data_block[511] = 0;
}

__syncthreads();

down_sweep_512((uint*) &data_block[0]);

// Store back elements
if (elem_index < array_size) {
dev_main_array[elem_index] = data_block[threadIdx.x];
}
if ((elem_index+blockDim.x) < array_size) {
dev_main_array[elem_index + blockDim.x] = data_block[threadIdx.x + blockDim.x];
}
}
}
}