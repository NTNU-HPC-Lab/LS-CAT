#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaKernel_BatchResize_GRAY2GRAY( int src_width, unsigned char* src_image, int num_rects, int* rects, int dst_width, int dst_height, float* dst_ptr )
{
const int gid = blockIdx.x * blockDim.x + threadIdx.x;
const int dst_image_size = dst_width * dst_height;
if( num_rects*dst_image_size <= gid ){
return;
}

const int image_index = (int)(gid / dst_image_size);
const int pixel_index = gid % dst_image_size;

float scale_x = (float)(rects[image_index*4 + 2])/dst_width;
float fx = (float)(((pixel_index % dst_width)+0.5f)*scale_x - 0.5);
int coor_x_in_rect = floor(fx);
fx = 1.0f - (fx - (float)coor_x_in_rect);

float scale_y = (float)(rects[image_index*4 + 3])/dst_height;
float fy = (float)(((pixel_index / dst_width)+0.5f)*scale_y - 0.5);
int coor_y_in_rect = floor(fy);
fy = 1.0f - (fy - (float)coor_y_in_rect);

int src_x = rects[image_index*4 + 0];
int src_y = rects[image_index*4 + 1];

float value = 0.;
value += (float)src_image[src_width*(src_y + coor_y_in_rect + 0) + (src_x + coor_x_in_rect + 0)] * fx * fy;
value += (float)src_image[src_width*(src_y + coor_y_in_rect + 0) + (src_x + coor_x_in_rect + 1)] * (1.0f - fx)*fy;
value += (float)src_image[src_width*(src_y + coor_y_in_rect + 1) + (src_x + coor_x_in_rect + 0)] * fx*(1.0f - fy);
value += (float)src_image[src_width*(src_y + coor_y_in_rect + 1) + (src_x + coor_x_in_rect + 1)] * (1.0f - fx)*(1.0f - fy);

dst_ptr[blockIdx.x * blockDim.x + threadIdx.x] = value / 255.f;
}