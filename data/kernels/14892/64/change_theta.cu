#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void change_theta(const int ncoord, const float3 *theta, float4 *thetax, float4 *thetay, float4 *thetaz) {

unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
if (pos < ncoord) {
thetax[pos].x = theta[pos*4].x;
thetax[pos].y = theta[pos*4+1].x;
thetax[pos].z = theta[pos*4+2].x;
thetax[pos].w = theta[pos*4+3].x;

thetay[pos].x = theta[pos*4].y;
thetay[pos].y = theta[pos*4+1].y;
thetay[pos].z = theta[pos*4+2].y;
thetay[pos].w = theta[pos*4+3].y;

thetaz[pos].x = theta[pos*4].z;
thetaz[pos].y = theta[pos*4+1].z;
thetaz[pos].z = theta[pos*4+2].z;
thetaz[pos].w = theta[pos*4+3].z;
}

}