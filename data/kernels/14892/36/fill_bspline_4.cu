#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_bspline_4(const float4 *xyzq, const int ncoord, const float *recip, const int nfftx, const int nffty, const int nfftz, int *gix, int *giy, int *giz, float *charge, float *thetax, float *thetay, float *thetaz, float *dthetax, float *dthetay, float *dthetaz) {

// Position to xyzq and atomgrid
unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;

while (pos < ncoord) {
float4 xyzqi = xyzq[pos];
float x = xyzqi.x;
float y = xyzqi.y;
float z = xyzqi.z;
float q = xyzqi.w;

float w;
// NOTE: I don't think we need the +2.0f here..
w = x*recip[0] + y*recip[1] + z*recip[2] + 2.0f;
float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));

w = x*recip[3] + y*recip[4] + z*recip[5] + 2.0f;
float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));

w = x*recip[6] + y*recip[7] + z*recip[8] + 2.0f;
float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

int frxi = (int)(frx);
int fryi = (int)(fry);
int frzi = (int)(frz);

float wx = frx - (float)frxi;
float wy = fry - (float)fryi;
float wz = frz - (float)frzi;

gix[pos] = frxi;
giy[pos] = fryi;
giz[pos] = frzi;
charge[pos] = q;

float3 theta_tmp[4];
float3 dtheta_tmp[4];

theta_tmp[3].x = 0.0f;
theta_tmp[3].y = 0.0f;
theta_tmp[3].z = 0.0f;
theta_tmp[1].x = wx;
theta_tmp[1].y = wy;
theta_tmp[1].z = wz;
theta_tmp[0].x = 1.0f - wx;
theta_tmp[0].y = 1.0f - wy;
theta_tmp[0].z = 1.0f - wz;

// compute standard b-spline recursion
theta_tmp[2].x = 0.5f*wx*theta_tmp[1].x;
theta_tmp[2].y = 0.5f*wy*theta_tmp[1].y;
theta_tmp[2].z = 0.5f*wz*theta_tmp[1].z;

theta_tmp[1].x = 0.5f*((wx+1.0f)*theta_tmp[0].x + (2.0f-wx)*theta_tmp[1].x);
theta_tmp[1].y = 0.5f*((wy+1.0f)*theta_tmp[0].y + (2.0f-wy)*theta_tmp[1].y);
theta_tmp[1].z = 0.5f*((wz+1.0f)*theta_tmp[0].z + (2.0f-wz)*theta_tmp[1].z);

theta_tmp[0].x = 0.5f*(1.0f-wx)*theta_tmp[0].x;
theta_tmp[0].y = 0.5f*(1.0f-wy)*theta_tmp[0].y;
theta_tmp[0].z = 0.5f*(1.0f-wz)*theta_tmp[0].z;

// perform standard b-spline differentiationa
dtheta_tmp[0].x = -theta_tmp[0].x;
dtheta_tmp[0].y = -theta_tmp[0].y;
dtheta_tmp[0].z = -theta_tmp[0].z;

dtheta_tmp[1].x = theta_tmp[0].x - theta_tmp[1].x;
dtheta_tmp[1].y = theta_tmp[0].y - theta_tmp[1].y;
dtheta_tmp[1].z = theta_tmp[0].z - theta_tmp[1].z;

dtheta_tmp[2].x = theta_tmp[1].x - theta_tmp[2].x;
dtheta_tmp[2].y = theta_tmp[1].y - theta_tmp[2].y;
dtheta_tmp[2].z = theta_tmp[1].z - theta_tmp[2].z;

dtheta_tmp[3].x = theta_tmp[2].x - theta_tmp[3].x;
dtheta_tmp[3].y = theta_tmp[2].y - theta_tmp[3].y;
dtheta_tmp[3].z = theta_tmp[2].z - theta_tmp[3].z;

// one more recursion
theta_tmp[3].x = (1.0f/3.0f)*wx*theta_tmp[2].x;
theta_tmp[3].y = (1.0f/3.0f)*wy*theta_tmp[2].y;
theta_tmp[3].z = (1.0f/3.0f)*wz*theta_tmp[2].z;

theta_tmp[2].x = (1.0f/3.0f)*((wx+1.0f)*theta_tmp[1].x + (3.0f-wx)*theta_tmp[2].x);
theta_tmp[2].y = (1.0f/3.0f)*((wy+1.0f)*theta_tmp[1].y + (3.0f-wy)*theta_tmp[2].y);
theta_tmp[2].z = (1.0f/3.0f)*((wz+1.0f)*theta_tmp[1].z + (3.0f-wz)*theta_tmp[2].z);

theta_tmp[1].x = (1.0f/3.0f)*((wx+2.0f)*theta_tmp[0].x + (2.0f-wx)*theta_tmp[1].x);
theta_tmp[1].y = (1.0f/3.0f)*((wy+2.0f)*theta_tmp[0].y + (2.0f-wy)*theta_tmp[1].y);
theta_tmp[1].z = (1.0f/3.0f)*((wz+2.0f)*theta_tmp[0].z + (2.0f-wz)*theta_tmp[1].z);

theta_tmp[0].x = (1.0f/3.0f)*(1.0f-wx)*theta_tmp[0].x;
theta_tmp[0].y = (1.0f/3.0f)*(1.0f-wy)*theta_tmp[0].y;
theta_tmp[0].z = (1.0f/3.0f)*(1.0f-wz)*theta_tmp[0].z;

// Store theta_tmp and dtheta_tmp into global memory
int pos4 = pos*4;
thetax[pos4]   = theta_tmp[0].x;
thetax[pos4+1] = theta_tmp[1].x;
thetax[pos4+2] = theta_tmp[2].x;
thetax[pos4+3] = theta_tmp[3].x;

thetay[pos4]   = theta_tmp[0].y;
thetay[pos4+1] = theta_tmp[1].y;
thetay[pos4+2] = theta_tmp[2].y;
thetay[pos4+3] = theta_tmp[3].y;

thetaz[pos4]   = theta_tmp[0].z;
thetaz[pos4+1] = theta_tmp[1].z;
thetaz[pos4+2] = theta_tmp[2].z;
thetaz[pos4+3] = theta_tmp[3].z;

dthetax[pos4]   = dtheta_tmp[0].x;
dthetax[pos4+1] = dtheta_tmp[1].x;
dthetax[pos4+2] = dtheta_tmp[2].x;
dthetax[pos4+3] = dtheta_tmp[3].x;

dthetay[pos4]   = dtheta_tmp[0].y;
dthetay[pos4+1] = dtheta_tmp[1].y;
dthetay[pos4+2] = dtheta_tmp[2].y;
dthetay[pos4+3] = dtheta_tmp[3].y;

dthetaz[pos4]   = dtheta_tmp[0].z;
dthetaz[pos4+1] = dtheta_tmp[1].z;
dthetaz[pos4+2] = dtheta_tmp[2].z;
dthetaz[pos4+3] = dtheta_tmp[3].z;

pos += blockDim.x*gridDim.x;
}

}