#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "fill_bspline_4.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float4 *xyzq = NULL;
hipMalloc(&xyzq, XSIZE*YSIZE);
const int ncoord = 1;
const float *recip = NULL;
hipMalloc(&recip, XSIZE*YSIZE);
const int nfftx = 1;
const int nffty = 1;
const int nfftz = 1;
int *gix = NULL;
hipMalloc(&gix, XSIZE*YSIZE);
int *giy = NULL;
hipMalloc(&giy, XSIZE*YSIZE);
int *giz = NULL;
hipMalloc(&giz, XSIZE*YSIZE);
float *charge = NULL;
hipMalloc(&charge, XSIZE*YSIZE);
float *thetax = NULL;
hipMalloc(&thetax, XSIZE*YSIZE);
float *thetay = NULL;
hipMalloc(&thetay, XSIZE*YSIZE);
float *thetaz = NULL;
hipMalloc(&thetaz, XSIZE*YSIZE);
float *dthetax = NULL;
hipMalloc(&dthetax, XSIZE*YSIZE);
float *dthetay = NULL;
hipMalloc(&dthetay, XSIZE*YSIZE);
float *dthetaz = NULL;
hipMalloc(&dthetaz, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
fill_bspline_4<<<gridBlock,threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
fill_bspline_4<<<gridBlock,threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
fill_bspline_4<<<gridBlock,threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}