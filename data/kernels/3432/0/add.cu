#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void add(long* a, long* b, long* c, long N) { //core from ScalarMultiplication_example1
long baseIdx = threadIdx.x;
long idx = baseIdx;
while (idx < N)
{
c[idx] = a[idx] * b[idx];
idx += blockDim.x;
}
__syncthreads();
long step = N / 2;
while (step != 0) {
idx = baseIdx;
while (idx < step) {
c[idx] += c[idx + step];
idx += blockDim.x;
}
step /= 2;
__syncthreads();
}
}