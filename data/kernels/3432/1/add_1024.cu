#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void add_1024(long* a, long* b, long* c, long N) { //more simple and probably faster core but works only with 1024 or less elements in vector in this example
c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
__syncthreads();
long step = N / 2;
while (step != 0) {
if (threadIdx.x < step)
{
c[threadIdx.x] += c[threadIdx.x + step];
}
step /= 2;
__syncthreads();
}
}