#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_image2D1C_ConvolveColumn(float* img, int n_x, int n_y, short k, float *kernel, float* out)
{
// Find index of current thread
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
if (idx_x>=n_x) return;
if (idx_y>=n_y) return;
float sum=0;
for (short i=-k;i<=k;i++)
{
short y=idx_y+i;
if (y<0) y=0;
if (y>=n_y) y=n_y-1;
sum+=kernel[i+k]*img[y*n_x+idx_x];
}
out[idx_y*n_x+idx_x]=sum;
}