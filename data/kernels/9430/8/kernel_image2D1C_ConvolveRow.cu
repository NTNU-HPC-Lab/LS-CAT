#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_image2D1C_ConvolveRow(float* img, int n_x, int n_y, short k, float *kernel, float* out)
{
// Find index of current thread
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
if (idx_x>=n_x) return;
if (idx_y>=n_y) return;

float sum=0;
for (short i=-k;i<=k;i++)
{
short x=idx_x+i;
if (x<0) x=0;
if (x>=n_x) x=n_x-1;
sum+=kernel[i+k]*img[idx_y*n_x+x];
}
out[idx_y*n_x+idx_x]=sum;
}