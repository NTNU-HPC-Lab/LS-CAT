#include "hip/hip_runtime.h"
#include "includes.h"

#define WEIGHTSUM 273
#define BLOCK_SIZE 16

int * heatmap;
size_t heatmap_pitch;

int * scaled_heatmap;
size_t scaled_heatmap_pitch;

int * blurred_heatmap;
size_t blurred_heatmap_pitch;

float* d_desiredPositionX;
float* d_desiredPositionY;

__global__ void computeHeatmap(float* desiredAgentsX, float* desiredAgentsY, int n, int* heatmap, size_t heatmap_pitch, int* scaled_heatmap, size_t scaled_heatmap_pitch) {
// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

// Thread row and column block
int row = threadIdx.y;
int col = threadIdx.x;

// x, y coordinate
int x = blockCol * blockDim.x + col;
int y = blockRow * blockDim.y + row;

// fade heatmap
int* heatPoint = (int*)((char*)heatmap + y * heatmap_pitch) + x;
*heatPoint = (int)round((*heatPoint) * 0.80);

// pull desiredAgentxX and Y array from global to shared memory, only 1 thread will do it
extern __shared__ float desiredPosition[];

if (row == 0 && col == 0) {
for (int i = 0; i < n; i++) {
desiredPosition[i] = desiredAgentsX[i];
desiredPosition[i + n] = desiredAgentsY[i];
}
}

__syncthreads();

// Count how many agents want to go to each location
for (int i = 0; i < n; i++) {
int desiredX = (int)desiredPosition[i];
int desiredY = (int)desiredPosition[i + n];

if (x == desiredX && y == desiredY) {
// intensify heat for better color results
if ((*heatPoint) + 40 <= 255) {
*heatPoint += 40;
}
}
}
}