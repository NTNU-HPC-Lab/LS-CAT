#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePositionParallel(float *agentsX, float *agentsY, float *destX, float *destY, float *destR, int n, int *reached) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int i = index; i < n; i += stride) {
// if there is no destination to go to
if (destX[i] == -1 || destY[i] == -1) {
continue;
}

// compute and update next position
double diffX = destX[i] - agentsX[i];
double diffY = destY[i] - agentsY[i];
double length = sqrtf(diffX * diffX + diffY * diffY);
agentsX[i] = (float)llrintf(agentsX[i] + diffX / length);
agentsY[i] = (float)llrintf(agentsY[i] + diffY / length);

// check if next position is inside the destination radius
diffX = destX[i] - agentsX[i];
diffY = destY[i] - agentsY[i];
length = sqrtf(diffX * diffX + diffY * diffY);

if (length < destR[i]) {
reached[i] = 1;
}
}
}