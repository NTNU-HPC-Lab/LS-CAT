#include "hip/hip_runtime.h"
#include "includes.h"

#define WEIGHTSUM 273
#define BLOCK_SIZE 16

int * heatmap;
size_t heatmap_pitch;

int * scaled_heatmap;
size_t scaled_heatmap_pitch;

int * blurred_heatmap;
size_t blurred_heatmap_pitch;

float* d_desiredPositionX;
float* d_desiredPositionY;

__global__ void computeScaledHeatmap(int* heatmap, size_t heatmap_pitch, int* scaled_heatmap, size_t scaled_heatmap_pitch) {
// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

// Thread row and column block
int row = threadIdx.y;
int col = threadIdx.x;

// x, y coordinate
int x = blockCol * blockDim.x + col;
int y = blockRow * blockDim.y + row;

// Scale the data for visual representation
int value = *((int*)((char*)heatmap + y * heatmap_pitch) + x);
for (int r = 0; r < CELLSIZE; r++) {
int* row = (int*)((char*)scaled_heatmap + (r + y * CELLSIZE) * scaled_heatmap_pitch);
for (int c = 0; c < CELLSIZE; c++) {
row[x * CELLSIZE + c] = value;
}
}
}