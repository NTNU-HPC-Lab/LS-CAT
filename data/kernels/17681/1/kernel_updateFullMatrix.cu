#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_updateFullMatrix( float * device_fullMatrix, float * B, float * V, float * Cm, float * Em, float * Rm, float dt, unsigned int nComp ) {
//TODO: fix memory usage matter

unsigned int t = threadIdx.x;
unsigned int baseIndex = t*nComp;

unsigned int i;
for ( i = 0; i < nComp; i++ )
{
unsigned int myIndex=baseIndex+i;
B[myIndex  ] =
V[ myIndex] * Cm[myIndex] 	/ ( dt / 2.0 ) +
Em[ myIndex] / Rm[myIndex];
}
__syncthreads();
}