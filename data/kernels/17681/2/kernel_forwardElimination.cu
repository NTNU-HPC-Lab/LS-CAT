#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_forwardElimination( float * fullMatrix, float * B, unsigned int nComp ) {
unsigned int t = threadIdx.x;
unsigned int baseIndex = t*nComp*nComp;

unsigned int i,j,k;
for ( i = 0; i < nComp - 1; i++ )
for ( j = i + 1; j < nComp; j++ ) {
double div = fullMatrix[baseIndex+ j*nComp+i ] / fullMatrix[baseIndex+ i*nComp+ i ];
for ( k = 0; k < nComp; k++ )
fullMatrix[ baseIndex+j*nComp+k ] -= div * fullMatrix[baseIndex+ i *nComp+ k ];
B[ baseIndex+j ] -= div * B[ baseIndex+i ];
}
__syncthreads();
}