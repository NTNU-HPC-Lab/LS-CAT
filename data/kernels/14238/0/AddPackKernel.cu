#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int GetVecIndex(int vecNumber, int dimCount, int *dimSizes, int measCount, int vecCount, int *dims)
{
unsigned long int index = 0;

for (int i = 0; i < dimCount; ++i)
index += (unsigned long int)dimSizes[i] * (unsigned long int)dims[i * vecCount + vecNumber];

return index;
}
__global__ void AddPackKernel(unsigned long int *codes, int *measures, int dimensionsCount, int *dimendionsSizes, int measuresCount, int currentCapacity, int fullCapacity, int packCount, int *packDimensions, int *packMeasures)
{
int currentVec = blockIdx.x * blockDim.x + threadIdx.x;

while (currentVec < packCount)
{
codes[currentCapacity + currentVec] = GetVecIndex(currentVec, dimensionsCount, dimendionsSizes, measuresCount, packCount, packDimensions);

for (int i = 0; i < measuresCount; ++i)
measures[i * fullCapacity + currentCapacity + currentVec] = packMeasures[i * packCount + currentVec];

currentVec += blockDim.x * gridDim.x;
}

}