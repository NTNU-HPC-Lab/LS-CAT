#include "hip/hip_runtime.h"
#include "includes.h"
//*************inclución de librerias***************


//************variables globales***************

int N=93, dimx=1920, dimy=2560, tam_imag=1920*2560;

//**********KERNEL**************

float *leerMatrizVarianza(int d);

//*****************función main**********************

__global__ void kernel (float *max, float *var, int *top, int k){
int idx=threadIdx.x + blockIdx.x*blockDim.x;
int tam_imag=1920*2560;

if(idx<tam_imag){
if(var[idx]>max[idx]){
top[idx]=k;
max[idx]=var[idx];
}
}
}