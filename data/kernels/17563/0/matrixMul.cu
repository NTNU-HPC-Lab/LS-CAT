#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int* A, int* B, int* C, int aF, int aC, int bF, int bC, int cF, int cC) {
// Compute each thread's global row and column index
int row = (blockIdx.y * blockDim.y) + threadIdx.y;
int col = (blockIdx.x * blockDim.x) + threadIdx.x;

// Iterate over row, and down column
////c[row * N + col] = 0;
if (aC != bF) return;
if ((row < aF) && (col < bC)) {
for (int k = 0; k < aC; ++k) {
// Accumulate results for a single element
C[row * cC + col] += A[row * aC + k] * B[k * bC + col];
}
}
//C[row * aF + col] = 0;
}