#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void detect_edges(unsigned char *in, unsigned char *out) {
int i;
int n_pixels = width * height;

for(i=0;i<n_pixels;i++) {
int x, y; // the pixel of interest
int b, d, f, h; // the pixels adjacent to x,y used for the calculation
int r; // the result of calculate

y = i / width;
x = i - (width * y);

if (x == 0 || y == 0 || x == width - 1 || y == height - 1) {
out[i] = 0;
} else {
b = i + width;
d = i - 1;
f = i + 1;
h = i - width;

r = (in[i] * 4) + (in[b] * -1) + (in[d] * -1) + (in[f] * -1)
+ (in[h] * -1);

if (r > 0) { // if the result is positive this is an edge pixel
out[i] = 255;
} else {
out[i] = 0;
}
}
}
}