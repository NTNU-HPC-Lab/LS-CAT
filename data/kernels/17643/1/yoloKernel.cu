#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float sigmoid(float data){ return 1./(1. + expf(-data)); };
__global__ void yoloKernel(const int n,const float * input, float* output, const int* anchors,int anchor_num, int classes,int height,int width,float down_stride,float thresh){
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx >= n) return;
extern __shared__ int shared_anchors[];
if(threadIdx.x < anchor_num*2){
shared_anchors[threadIdx.x] = anchors[threadIdx.x];
}
__syncthreads();
int row = idx % width;
int col = (idx / width) % height;
int anchor_id = (idx / width / height)% anchor_num;
int batch_id = idx/width/height/anchor_num;
int C = anchor_num*(classes+5);
int stride = width*height;
int begin_id =  ((batch_id * C + anchor_id*(classes + 5))*height+col)*width+row;
float conf_prob =sigmoid(input[begin_id + 4*stride]);
if(conf_prob > thresh) {
int class_id = -1;
float max_prob = thresh;
for (int c = 0;c<classes;++c){
int cls_id = begin_id + stride*(c + 5);
float cls_prob =  sigmoid(input[cls_id]) *conf_prob ;
if(cls_prob > max_prob){
max_prob = cls_prob;
class_id = c;
}
}
if(class_id >= 0){
int resCount = (int)atomicAdd(output,1);
float * data = output + 1 + resCount*7;
// x1,y1,x2,y2,cls,conf,batch_id
data[0] = (row + sigmoid(input[begin_id]))*down_stride;
data[1] = (col  + sigmoid(input[begin_id+stride]))*down_stride;
data[2] = expf(input[begin_id+2*stride]) * (float)shared_anchors[2*anchor_id];
data[3] = expf(input[begin_id+3*stride]) * (float)shared_anchors[2*anchor_id + 1];
data[4] = class_id;
data[5] = max_prob;
data[6] = batch_id;
}
}
}