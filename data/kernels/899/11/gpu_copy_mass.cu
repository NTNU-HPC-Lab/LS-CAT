#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copy_mass( const int num_atoms, const int* g_group_contents, const double* g_mass_i, double* g_mass_o)
{
const int n = threadIdx.x + blockIdx.x * blockDim.x;
if (n < num_atoms) {
g_mass_o[n] = g_mass_i[g_group_contents[n]];
}
}