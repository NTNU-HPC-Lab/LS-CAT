#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_find_hac(const int Nc, const int Nd, const double* g_heat, double* g_hac)
{
//<<<Nc, 128>>>

__shared__ double s_hac_xi[128];
__shared__ double s_hac_xo[128];
__shared__ double s_hac_yi[128];
__shared__ double s_hac_yo[128];
__shared__ double s_hac_z[128];

int tid = threadIdx.x;
int bid = blockIdx.x;
int number_of_patches = (Nd - 1) / 128 + 1;
int number_of_data = Nd - bid;

s_hac_xi[tid] = 0.0;
s_hac_xo[tid] = 0.0;
s_hac_yi[tid] = 0.0;
s_hac_yo[tid] = 0.0;
s_hac_z[tid] = 0.0;

for (int patch = 0; patch < number_of_patches; ++patch) {
int index = tid + patch * 128;
if (index + bid < Nd) {
s_hac_xi[tid] += g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 0] +
g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 1];
s_hac_xo[tid] += g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 1] +
g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 0];
s_hac_yi[tid] += g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 2] +
g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 3];
s_hac_yo[tid] += g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 3] +
g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 2];
s_hac_z[tid] += g_heat[index + Nd * 4] * g_heat[index + bid + Nd * 4];
}
}
__syncthreads();

#pragma unroll
for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
if (tid < offset) {
s_hac_xi[tid] += s_hac_xi[tid + offset];
s_hac_xo[tid] += s_hac_xo[tid + offset];
s_hac_yi[tid] += s_hac_yi[tid + offset];
s_hac_yo[tid] += s_hac_yo[tid + offset];
s_hac_z[tid] += s_hac_z[tid + offset];
}
__syncthreads();
}

if (tid == 0) {
g_hac[bid + Nc * 0] = s_hac_xi[0] / number_of_data;
g_hac[bid + Nc * 1] = s_hac_xo[0] / number_of_data;
g_hac[bid + Nc * 2] = s_hac_yi[0] / number_of_data;
g_hac[bid + Nc * 3] = s_hac_yo[0] / number_of_data;
g_hac[bid + Nc * 4] = s_hac_z[0] / number_of_data;
}
}