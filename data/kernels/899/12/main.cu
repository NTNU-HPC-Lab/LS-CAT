#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "gpu_copy_velocity.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int num_atoms = 1;
const int *g_group_contents = NULL;
hipMalloc(&g_group_contents, XSIZE*YSIZE);
const double *g_vx_i = NULL;
hipMalloc(&g_vx_i, XSIZE*YSIZE);
const double *g_vy_i = NULL;
hipMalloc(&g_vy_i, XSIZE*YSIZE);
const double *g_vz_i = NULL;
hipMalloc(&g_vz_i, XSIZE*YSIZE);
double *g_vx_o = NULL;
hipMalloc(&g_vx_o, XSIZE*YSIZE);
double *g_vy_o = NULL;
hipMalloc(&g_vy_o, XSIZE*YSIZE);
double *g_vz_o = NULL;
hipMalloc(&g_vz_o, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gpu_copy_velocity<<<gridBlock,threadBlock>>>(num_atoms,g_group_contents,g_vx_i,g_vy_i,g_vz_i,g_vx_o,g_vy_o,g_vz_o);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
gpu_copy_velocity<<<gridBlock,threadBlock>>>(num_atoms,g_group_contents,g_vx_i,g_vy_i,g_vz_i,g_vx_o,g_vy_o,g_vz_o);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
gpu_copy_velocity<<<gridBlock,threadBlock>>>(num_atoms,g_group_contents,g_vx_i,g_vy_i,g_vz_i,g_vx_o,g_vy_o,g_vz_o);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}