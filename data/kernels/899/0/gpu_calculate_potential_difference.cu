#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_calculate_potential_difference( const int size, const int number_of_rounds, const double* potential_per_atom, const double* potential_per_atom_temp, double* potential_difference)
{
__shared__ double s_diff[1024];
s_diff[threadIdx.x] = 0.0;

double diff = 0.0f;

for (int round = 0; round < number_of_rounds; ++round) {
const int n = threadIdx.x + round * 1024;
if (n < size) {
diff += potential_per_atom_temp[n] - potential_per_atom[n];
}
}

s_diff[threadIdx.x] = diff;
__syncthreads();

for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
if (threadIdx.x < offset) {
s_diff[threadIdx.x] += s_diff[threadIdx.x + offset];
}
__syncthreads();
}

if (threadIdx.x == 0) {
potential_difference[0] = s_diff[0];
}
}