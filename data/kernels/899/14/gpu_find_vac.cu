#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_find_vac( const int num_atoms, const int correlation_step, const double* g_mass, const double* g_vx, const double* g_vy, const double* g_vz, const double* g_vx_all, const double* g_vy_all, const double* g_vz_all, double* g_vac_x, double* g_vac_y, double* g_vac_z)
{
int tid = threadIdx.x;
int bid = blockIdx.x;
int size_sum = bid * num_atoms;
int number_of_rounds = (num_atoms - 1) / 128 + 1;
__shared__ double s_vac_x[128];
__shared__ double s_vac_y[128];
__shared__ double s_vac_z[128];
double vac_x = 0.0;
double vac_y = 0.0;
double vac_z = 0.0;

for (int round = 0; round < number_of_rounds; ++round) {
int n = tid + round * 128;
if (n < num_atoms) {
double mass = g_mass[n];
vac_x += mass * g_vx[n] * g_vx_all[size_sum + n];
vac_y += mass * g_vy[n] * g_vy_all[size_sum + n];
vac_z += mass * g_vz[n] * g_vz_all[size_sum + n];
}
}
s_vac_x[tid] = vac_x;
s_vac_y[tid] = vac_y;
s_vac_z[tid] = vac_z;
__syncthreads();

for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
if (tid < offset) {
s_vac_x[tid] += s_vac_x[tid + offset];
s_vac_y[tid] += s_vac_y[tid + offset];
s_vac_z[tid] += s_vac_z[tid + offset];
}
__syncthreads();
}

if (tid == 0) {
if (bid <= correlation_step) {
g_vac_x[correlation_step - bid] += s_vac_x[0];
g_vac_y[correlation_step - bid] += s_vac_y[0];
g_vac_z[correlation_step - bid] += s_vac_z[0];
} else {
g_vac_x[correlation_step + gridDim.x - bid] += s_vac_x[0];
g_vac_y[correlation_step + gridDim.x - bid] += s_vac_y[0];
g_vac_z[correlation_step + gridDim.x - bid] += s_vac_z[0];
}
}
}