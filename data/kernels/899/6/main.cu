#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "gpu_find_vac.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int num_atoms = 1;
const int correlation_step = 1;
const double *g_vx = NULL;
hipMalloc(&g_vx, XSIZE*YSIZE);
const double *g_vy = NULL;
hipMalloc(&g_vy, XSIZE*YSIZE);
const double *g_vz = NULL;
hipMalloc(&g_vz, XSIZE*YSIZE);
const double *g_vx_all = NULL;
hipMalloc(&g_vx_all, XSIZE*YSIZE);
const double *g_vy_all = NULL;
hipMalloc(&g_vy_all, XSIZE*YSIZE);
const double *g_vz_all = NULL;
hipMalloc(&g_vz_all, XSIZE*YSIZE);
double *g_vac_x = NULL;
hipMalloc(&g_vac_x, XSIZE*YSIZE);
double *g_vac_y = NULL;
hipMalloc(&g_vac_y, XSIZE*YSIZE);
double *g_vac_z = NULL;
hipMalloc(&g_vac_z, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gpu_find_vac<<<gridBlock,threadBlock>>>(num_atoms,correlation_step,g_vx,g_vy,g_vz,g_vx_all,g_vy_all,g_vz_all,g_vac_x,g_vac_y,g_vac_z);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
gpu_find_vac<<<gridBlock,threadBlock>>>(num_atoms,correlation_step,g_vx,g_vy,g_vz,g_vx_all,g_vy_all,g_vz_all,g_vac_x,g_vac_y,g_vac_z);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
gpu_find_vac<<<gridBlock,threadBlock>>>(num_atoms,correlation_step,g_vx,g_vy,g_vz,g_vx_all,g_vy_all,g_vz_all,g_vac_x,g_vac_y,g_vac_z);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}