#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copy_velocity( const int num_atoms, const int offset, const int* g_group_contents, const double* g_vx_i, const double* g_vy_i, const double* g_vz_i, float* g_vx_o, float* g_vy_o, float* g_vz_o)
{
const int n = threadIdx.x + blockIdx.x * blockDim.x;
if (n < num_atoms) {
const int m = g_group_contents[offset + n];
g_vx_o[n] = g_vx_i[m];
g_vy_o[n] = g_vy_i[m];
g_vz_o[n] = g_vz_i[m];
}
}