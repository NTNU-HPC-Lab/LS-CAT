#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copy_velocity( const int num_atoms, const double* g_vx_i, const double* g_vy_i, const double* g_vz_i, double* g_vx_o, double* g_vy_o, double* g_vz_o)
{
const int n = threadIdx.x + blockIdx.x * blockDim.x;
if (n < num_atoms) {
g_vx_o[n] = g_vx_i[n];
g_vy_o[n] = g_vy_i[n];
g_vz_o[n] = g_vz_i[n];
}
}