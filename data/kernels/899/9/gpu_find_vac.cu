#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_find_vac( const int num_atoms, const int correlation_step, const int num_correlation_steps, const float* g_vx, const float* g_vy, const float* g_vz, const float* g_vx_all, const float* g_vy_all, const float* g_vz_all, float* g_vac_x, float* g_vac_y, float* g_vac_z)
{
const int num_atoms_sq = num_atoms * num_atoms;
const int n1n2 = blockIdx.x * blockDim.x + threadIdx.x;
if (n1n2 >= num_atoms_sq)
return;
const int n1 = n1n2 / num_atoms;
const int n2 = n1n2 - n1 * num_atoms;
for (int k = 0; k < num_correlation_steps; ++k) {
int nc = correlation_step - k;
if (nc < 0)
nc += num_correlation_steps;
g_vac_x[nc * num_atoms_sq + n1n2] += g_vx[n1] * g_vx_all[k * num_atoms + n2];
g_vac_y[nc * num_atoms_sq + n1n2] += g_vy[n1] * g_vy_all[k * num_atoms + n2];
g_vac_z[nc * num_atoms_sq + n1n2] += g_vz[n1] * g_vz_all[k * num_atoms + n2];
}
}