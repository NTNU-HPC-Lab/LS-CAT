#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_positions( const int size, const double position_step, const double* force_per_atom, const double* position_per_atom, double* position_per_atom_temp)
{
const int n = blockIdx.x * blockDim.x + threadIdx.x;
if (n < size) {
const double position_change = force_per_atom[n] * position_step;
position_per_atom_temp[n] = position_per_atom[n] + position_change;
}
}