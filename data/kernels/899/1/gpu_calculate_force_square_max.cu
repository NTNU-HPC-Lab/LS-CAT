#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_calculate_force_square_max( const int size, const int number_of_rounds, const double* force_per_atom, double* force_square_max)
{
const int tid = threadIdx.x;

__shared__ double s_force_square[1024];
s_force_square[tid] = 0.0;

double force_square = 0.0;

for (int round = 0; round < number_of_rounds; ++round) {
const int n = tid + round * 1024;
if (n < size) {
const double f = force_per_atom[n];
if (f * f > force_square)
force_square = f * f;
}
}

s_force_square[tid] = force_square;
__syncthreads();

for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
if (tid < offset) {
if (s_force_square[tid + offset] > s_force_square[tid]) {
s_force_square[tid] = s_force_square[tid + offset];
}
}
__syncthreads();
}

if (tid == 0) {
force_square_max[0] = s_force_square[0];
}
}