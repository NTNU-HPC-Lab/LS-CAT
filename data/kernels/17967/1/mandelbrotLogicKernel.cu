#include "hip/hip_runtime.h"
#include "includes.h"



struct hipGraphicsResource* cuda_vbo_resource;


__global__ void mandelbrotLogicKernel(float* data, int width, int height, const int maxIteration, const double middlea, const double middleb, const double rangea, const double rangeb)
{
unsigned int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y_dim = blockIdx.y * blockDim.y + threadIdx.y;

int ipos = width * y_dim * 3 + x_dim * 3;

double x0 = (double)x_dim / (double)width;
double y0 = (double)y_dim / (double)height;

x0 = x0 * rangea + middlea - rangea / 2;
y0 = y0 * rangeb + middleb - rangeb / 2;

double real = 0;
double imaginary = 0;

int iteration = 0;
while (real * real + imaginary * imaginary <= 4 && iteration < maxIteration)
{
double temp = real * real - imaginary * imaginary + x0;
imaginary = 2 * real * imaginary + y0;
real = temp;
iteration++;
}

// Color algorithm from my brother (https://github.com/Julian-Wollersberger/Apfelmannchen)
int runde;
double fraction;

if (iteration == maxIteration)
{
data[ipos + 0] = 1.0f;
data[ipos + 1] = 1.0f;
data[ipos + 2] = 1.0f;
}
else
{
iteration += 8;
runde = 15;
while (iteration >= runde)
runde = (runde * 2) + 1;

fraction = (iteration - runde / 2) / (double)(runde / 2);

if (fraction < 0)
{
data[ipos + 0] = 1.0f;
data[ipos + 1] = 1.0f;
data[ipos + 2] = 1.0f;
}
else if (fraction < 1.0 / 6) { data[ipos + 0] = 1.0f;  data[ipos + 1] = 0.0f;  data[ipos + 2] = fraction * 6.0f; }
else if (fraction < 2.0 / 6) { data[ipos + 0] = 1 - (fraction - 1.0 / 6) * 6;  data[ipos + 1] = 0.0f;  data[ipos + 2] = 1.0f; }
else if (fraction < 3.0 / 6) { data[ipos + 0] = 0.0f;  data[ipos + 1] = (fraction - 2.0 / 6) * 6;  data[ipos + 2] = 1.0f; }
else if (fraction < 4.0 / 6) { data[ipos + 0] = 0.0f;  data[ipos + 1] = 1.0f;  data[ipos + 2] = 1 - (fraction - 3.0 / 6) * 6; }
else if (fraction < 5.0 / 6) { data[ipos + 0] = (fraction - 4.0 / 6) * 6;  data[ipos + 1] = 1.0f;  data[ipos + 2] = 0.0f; }
else if (fraction <= 6.0 / 6) { data[ipos + 0] = 1.0f;  data[ipos + 1] = 1 - (fraction - 5.0 / 6) * 6;  data[ipos + 2] = 0.0f; }
else
{
data[ipos + 0] = 0.0f;
data[ipos + 1] = 0.0f;
data[ipos + 2] = 0.0f;
}
}
}