#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_2_shared(int columns, int rows, float* mat1, float* matanswer) {
__shared__ float temp_answer[32];
if (threadIdx.y == 0) {
temp_answer[threadIdx.x] = float(0);
}
__syncthreads();  //Existe pero no es reconocido por Itellisense

int thread_mat_colid = blockIdx.x * blockDim.x + threadIdx.x;
int thread_mat_rowid = blockIdx.y * blockDim.y + threadIdx.y;
int position_in_matrix = thread_mat_rowid * columns + thread_mat_colid;

atomicAdd(&(temp_answer[threadIdx.x]), mat1[position_in_matrix]);
__syncthreads();  //Existe pero no es reconocido por Itellisense

if (blockIdx.x == 1 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
printf("%d Value %f \n", position_in_matrix, mat1[position_in_matrix]);
}

if (threadIdx.y == rows - 1) {
atomicAdd(&(matanswer[thread_mat_colid]), temp_answer[threadIdx.x]);
}
__syncthreads();
//printf("BlockID_x: %d BlockID_y: %d Blockdim_x: %d  Blockdim_y: %d ThreadIdx_x: %d ThreadIdx_y: %d \n",  blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
}