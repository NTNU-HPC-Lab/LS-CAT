#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_1(int columns, int rows, float* mat1, float* matanswer) {
int columna = threadIdx.x;  //En que columna operamos (no filas)
float temp_value = 0;

for (int k = 0; k < rows; k++) {
temp_value = temp_value + mat1[(k * columns) + columna];
}
matanswer[columna] = temp_value;
}