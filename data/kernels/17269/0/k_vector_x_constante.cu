#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void print_function(int*, int);

// Programando Tarjeta//


__global__ void k_vector_x_constante(int* arr, int* answer, int n, int k) { // arr -> Vector, n -> tamaño de array, k flotante
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < n) {
answer[idx] = arr[idx] * k;
}
}