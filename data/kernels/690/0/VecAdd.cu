#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void VecAdd(float * A, float * B, float * C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
C[i] = A[i] + B[i];
}