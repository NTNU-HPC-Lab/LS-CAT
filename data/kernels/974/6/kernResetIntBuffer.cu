#include "hip/hip_runtime.h"
#include "includes.h"
#define GLM_FORCE_CUDA

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x;
if (index < N) {
intBuffer[index] = value;
}
}