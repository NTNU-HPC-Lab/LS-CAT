#include "hip/hip_runtime.h"
#include "includes.h"
#define GLM_FORCE_CUDA

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices, int *gridCellStartIndices, int *gridCellEndIndices) {
// TODO-2.1
// Identify the start point of each cell in the gridIndices array.
// This is basically a parallel unrolling of a loop that goes
// "this index doesn't match the one before it, must be a new cell!"
}