#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_cuda_add_bias_kernel(float *output, float *bias, int num_channels, int spatial_size)
{
int offset = blockIdx.x * blockDim.x + threadIdx.x;
int channel = blockIdx.y;
int batch_size = blockIdx.z;

if (offset < spatial_size)
output[(batch_size * num_channels + channel) * spatial_size + offset] += bias[channel];
}