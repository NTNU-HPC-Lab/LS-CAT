#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *input, float *output, int *indexes)
{
int h = (in_h-1)/stride + 1;
int w = (in_w-1)/stride + 1;
int c = in_c;

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) {
return;
}

int j = id % w;
id /= w;
int i = id % h;
id /= h;
int k = id % c;
id /= c;
int b = id;

int out_index = j + w*(i + h*(k + c*b));
float max = -INFINITY;
int max_i = -1;
int l, m;
for (l = 0; l < size; ++l) {
for (m = 0; m < size; ++m) {
int cur_h = i * stride + l;
int cur_w = j * stride + m;
int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
int valid = (cur_h >= 0 && cur_h < in_h &&
cur_w >= 0 && cur_w < in_w);
float val = (valid != 0) ? input[index] : -INFINITY;
max_i = (val > max) ? index : max_i;
max   = (val > max) ? val   : max;
}
}
output[out_index] = max;
indexes[out_index] = max_i;
}