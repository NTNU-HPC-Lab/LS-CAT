#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_tanh_grad_kernel(int n, float *x, float *dx)
{
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
dx[i] *= (1 - x[i] * x[i]);
}
return;
}