#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _norm_forward_kernel(float *x, float *mean, float *variance, int b, int c, int wxh)
{
int ind = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
int j = (ind / wxh) % c;

if (ind >= b * c * wxh)
return;

x[ind] = (x[ind] - mean[j]) / (sqrt(variance[j] + 0.000001f));
}