#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_forward_softmax_layer_kernel(int n, int batch, float *input, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if (b >= batch) {
return;
}

for (i = 0; i < n; ++i) {
int val = input[i+b*n];
largest = (val>largest) ? val : largest;
}

for (i = 0; i < n; ++i) {
sum += exp(input[i+b*n]-largest);
}

sum = (sum != 0) ? largest+log(sum) : largest-100;

for (i = 0; i < n; ++i) {
output[i+b*n] = exp(input[i+b*n]-sum);
}
}