#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_vsum_kernel(int n, float *x, float *sum)
{
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n)
*sum += x[i];
}