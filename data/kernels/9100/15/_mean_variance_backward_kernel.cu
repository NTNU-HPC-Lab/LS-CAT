#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mean_variance_backward_kernel(float *x, float *grad, float *mean, float *var, int b, int c, int wxh, float *mean_diff, float *var_diff)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x, j, k, ind;

if (i >= c)
return;

mean_diff[i] = 0;
var_diff[i] = 0;
for (j = 0; j < b; ++j) {
for (k = 0; k < wxh; ++k) {
ind = j * c * wxh + i * wxh + k;
mean_diff[i] += grad[ind];
var_diff[i] += grad[ind] * (x[ind] - mean[i]);
}
}
mean_diff[i] *= (-1.0f / sqrt (var[i] + 0.00001f));
var_diff[i] *= -0.5f / (var[i] * sqrtf(var[i]) + 0.00001f);
}