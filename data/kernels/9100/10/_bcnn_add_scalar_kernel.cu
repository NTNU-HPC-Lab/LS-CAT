#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_add_scalar_kernel(int n, float a, float *y)
{
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n)
y[i] += a;
}