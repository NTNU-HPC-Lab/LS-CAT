#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_cuda_im2col_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col)
{
int i, j, w, h, w_out, h_index, h_out, channel_in, channel_out;
int h_in, w_in;
int index = blockIdx.x * blockDim.x + threadIdx.x;
float *data_col_ptr = NULL;
const float *data_im_ptr = NULL;

for(; index < n; index += blockDim.x * gridDim.x) {
w_out = index % width_col;
h_index = index / width_col;
h_out = h_index % height_col;
channel_in = h_index / height_col;
channel_out = channel_in * ksize * ksize;
h_in = h_out * stride - pad;
w_in = w_out * stride - pad;
data_col_ptr = data_col;
data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
data_im_ptr = data_im;
data_im_ptr += (channel_in * height + h_in) * width + w_in;
for (i = 0; i < ksize; ++i) {
for (j = 0; j < ksize; ++j) {
h = h_in + i;
w = w_in + j;
*data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
data_im_ptr[i * width + j] : 0;
data_col_ptr += height_col * width_col;
}
}
}
}