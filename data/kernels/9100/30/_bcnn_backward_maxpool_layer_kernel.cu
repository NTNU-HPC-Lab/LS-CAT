#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *diff, float *prev_delta, int *indexes)
{
int h = (in_h-1)/stride + 1;
int w = (in_w-1)/stride + 1;
int c = in_c;
int area = (size-1)/stride;

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= n) {
return;
}

int index = id;
int j = id % in_w;
id /= in_w;
int i = id % in_h;
id /= in_h;
int k = id % in_c;
id /= in_c;
int b = id;

int w_offset = (-size-1)/2 + 1;
int h_offset = (-size-1)/2 + 1;

float d = 0;
int l, m;
for (l = -area; l < area + 1; ++l) {
for (m = -area; m < area + 1; ++m) {
int out_w = (j - w_offset) / stride + m;
int out_h = (i - h_offset) / stride + l;
int out_index = out_w + w * (out_h + h * (k + c * b));
int valid = (out_w >= 0 && out_w < w &&
out_h >= 0 && out_h < h);
d += (valid && indexes[out_index] == index) ? diff[out_index] : 0;
}
}
prev_delta[index] += d;
}