#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_tanh_kernel(int n, float *x, float *y)
{
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
y[i] = (exp(2 * x[i]) - 1) / (exp(2 * x[i]) + 1);
}
return;
}