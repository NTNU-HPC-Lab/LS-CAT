#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_backward_depthwise_sep_conv_data_kernel(int nthreads, float *dst_grad, float *weight_data, int batch_size, const int channels, int dst_h, int dst_w, const int src_h, const int src_w, int kernel_sz, int stride, int pad, float *src_grad)
{

int i, n, c, h, w, kw, kh, h_out_s, w_out_s, h_out, w_out, offset;
float value = 0.0f;
float *weight = NULL;

for (i = blockIdx.x * blockDim.x + threadIdx.x; i < nthreads; i += blockDim.x * gridDim.x) {
n = i / channels / src_h / src_w;
c = (i / src_h / src_w) % channels;
h = (i / src_w) % src_h;
w = i % src_w;
weight = weight_data + c * kernel_sz * kernel_sz;
value = 0.0f;
for (kh = 0; kh < kernel_sz; ++kh) {
for (kw = 0; kw < kernel_sz; ++kw) {
h_out_s = h + pad - kh;
w_out_s = w + pad - kw;
if (((h_out_s % stride) == 0) && ((w_out_s % stride) == 0)) {
h_out = h_out_s / stride;
w_out = w_out_s / stride;
if ((h_out >= 0) && (h_out < dst_h) && (w_out >= 0) && (w_out < dst_w)) {
offset = ((n * channels + c) * dst_h + h_out) * dst_w + w_out;
value += (*weight) * dst_grad[offset];
}
}
++weight;
}
}
src_grad[i] += value;
}
}