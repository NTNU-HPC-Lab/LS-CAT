#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "_bcnn_backward_depthwise_sep_conv_data_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int nthreads = 1;
float *dst_grad = NULL;
hipMalloc(&dst_grad, XSIZE*YSIZE);
float *weight_data = NULL;
hipMalloc(&weight_data, XSIZE*YSIZE);
int batch_size = XSIZE*YSIZE;
const int channels = 1;
int dst_h = 1;
int dst_w = 1;
const int src_h = 1;
const int src_w = 1;
int kernel_sz = 1;
int stride = 2;
int pad = 2;
float *src_grad = NULL;
hipMalloc(&src_grad, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
_bcnn_backward_depthwise_sep_conv_data_kernel<<<gridBlock,threadBlock>>>(nthreads,dst_grad,weight_data,batch_size,channels,dst_h,dst_w,src_h,src_w,kernel_sz,stride,pad,src_grad);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
_bcnn_backward_depthwise_sep_conv_data_kernel<<<gridBlock,threadBlock>>>(nthreads,dst_grad,weight_data,batch_size,channels,dst_h,dst_w,src_h,src_w,kernel_sz,stride,pad,src_grad);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
_bcnn_backward_depthwise_sep_conv_data_kernel<<<gridBlock,threadBlock>>>(nthreads,dst_grad,weight_data,batch_size,channels,dst_h,dst_w,src_h,src_w,kernel_sz,stride,pad,src_grad);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}