#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_clamp_grad_kernel(int n, float *x, float *dx)
{
int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
if (i < n) {
dx[i] *= (float)(x[i] > 0.0f && (x[i] < 1.0f));
}
return;
}