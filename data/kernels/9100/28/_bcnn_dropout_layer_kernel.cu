#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_dropout_layer_kernel(float *input, int size, float *rand, float prob, float scale)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id < size) {
input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
}
}