#include "hip/hip_runtime.h"
#include "includes.h"

#define N 1200
#define THREADS 1024


__global__ void matrixMultKernel (double *a, double *b, double *c, int n)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

if((row < n) && (col < n)){
double v = 0;
for(int k = 0; k < n; k++){
v += a[row * n + k] * b[k * n + col];
}
c[row * n + col] = v;
}
}