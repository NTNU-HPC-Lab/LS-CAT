#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd (int *a, int *b, int *c)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < N){
c[index] = a[index] + b[index];
}
}