#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setDiffVolumeKernel(float *d_fv, unsigned char *d_picture1, unsigned char *d_picture2, unsigned picWidth, unsigned picHeight) {
__shared__ float p1_section[10 * 10 * 4];
__shared__ float p2_section[10 * 10 * 4];
unsigned i;

// This thread's position in its block's subsection of the float volume
unsigned sx, sy, sz;
// Dimensions of the grid
unsigned gx, gy, gz;
// Position of this thread's block
unsigned bx, by, bz;
// This thread's position in the entire float volume
unsigned vx, vy, vz;
// The location of the colors that this thread will be comparing
unsigned c1, c2;

// Get the position of this thread in its subsection
sz = threadIdx.x % 10;
sy = threadIdx.x / 100;
sx = (threadIdx.x % 100) / 10;

// Get the dimensions of the grid
gz = picWidth / 10;
if(picWidth % 10) gz++;
gy = picHeight / 10;
if(picHeight % 10) gy++;
gx = picWidth / 10;
if(picWidth % 10) gx++;

// Get the position of this thread's block
bz = blockIdx.x % gz;
by = blockIdx.x / (gx * gz);
bx = (blockIdx.x % (gx * gz)) / gz;

// Get the position of this thread in entire float volume
vx = sx + 10 * bx;
vy = sy + 10 * by;
vz = sz + 10 * bz;

// Copy subpicture to shared memory

// See if this thread needs to copy from picture 1
// picture 1 covers width * height

// If the float volume z of this thread is zero,
// then it needs to copy from picture 1
if(sz == 0) {
// Check if this thread will get a pixel not in the picture
if(vx < picWidth && vy < picHeight) {
for(i = 0; i < 4; i++) {
p1_section[(sx + sy * 10) * 4 + i] =
(float) d_picture1[(vx + vy * picWidth) * 4 + i];
}
}
}

// See if this thread needs to copy from picture 2
// picture 2 covers depth * height

// If the float volume x of this thread is zero,
// then it needs to copy from picture 2
if(sx == 0) {
// Check if this thread will get a pixel not in the picture
if(vz < picWidth && vy < picHeight) {
for(i = 0; i < 4; i++) {
p2_section[(sz + sy * 10) * 4 + i] =
(float) d_picture2[(vz + vy * picWidth) * 4 + i];
}
}
}

__syncthreads();
// Now each of d_picture1 and d_picture2 are properly filled out

// Write difference into float volume
if(vx < picWidth && vy < picHeight && vz < picWidth) {
c1 = (sx + sy * 10) * 4;
c2 = (sz + sy * 10) * 4;
d_fv[vz + vx * picWidth + vy * picWidth * picWidth] =
sqrtf(
powf(p1_section[c1 + 0] - p2_section[c2 + 0], 2.f) +
powf(p1_section[c1 + 1] - p2_section[c2 + 1], 2.f) +
powf(p1_section[c1 + 2] - p2_section[c2 + 2], 2.f) +
powf(p1_section[c1 + 3] - p2_section[c2 + 3], 2.f)
);
}
}