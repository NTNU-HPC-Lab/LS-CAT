#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to_float(float *out, int *in, int size) {
int element = threadIdx.x + blockDim.x * blockIdx.x;
if (element >= size) return;
out[element] = float(in[element]);
}