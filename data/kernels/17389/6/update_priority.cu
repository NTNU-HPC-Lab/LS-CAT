#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_priority(int *new_priority, int n_selected, const int *new_idx, int n_ws, const int *idx, const int *priority) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < n_selected) {
int my_new_idx = new_idx[tid];
// The working set size is limited (~1024 elements) so we just loop through it
for (int i = 0; i < n_ws; i++) {
if (idx[i] == my_new_idx) new_priority[tid] = priority[i] + 1;
}
}
}