#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void build_expected_output(int *output, int n_rows, int k, const int *labels) {
int row = threadIdx.x + blockDim.x * blockIdx.x;
if (row >= n_rows) return;

int cur_label = labels[row];
for (int i = 0; i < k; i++) {
output[row * k + i] = cur_label;
}
}