#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naiveHistKernel(int* bins, int nbins, int* in, int nrows) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
auto offset = blockIdx.y * nrows;
auto binOffset = blockIdx.y * nbins;
for (; tid < nrows; tid += stride) {
int id = in[offset + tid];
if (id < 0)
id = 0;
else if (id >= nbins)
id = nbins - 1;
in[offset + tid] = id;
atomicAdd(bins + binOffset + id, 1);
}
}