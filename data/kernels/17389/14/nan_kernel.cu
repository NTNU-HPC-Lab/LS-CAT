#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nan_kernel(float* data, const bool* mask, int len, float nan) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid >= len) return;
if (!mask[tid]) data[tid] = nan;
}