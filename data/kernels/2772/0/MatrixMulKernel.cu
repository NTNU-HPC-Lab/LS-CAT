#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 7

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

//Identify the row and column of the Pd element to work on
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;

//printf("Row : %d,  Col : %d\n", Row, Col);

float Pvalue = 0;
//Loop over the Md and Nd tiles required to compute the Pd element
for (int m = 0; m < Width / TILE_WIDTH; m++)
{
//Collaborative loading of Md and Nd tiles into shared memory
Mds[ty][tx] = Md[Row * Width + (m * TILE_WIDTH + tx)];
Nds[ty][tx] = Nd[(m * TILE_WIDTH + ty) * Width + Col];
__syncthreads();

for (int k = 0; k < TILE_WIDTH; k++)
Pvalue += Mds[ty][k] * Nds[k][tx];
__syncthreads();
}

Pd[Row * Width + Col] = Pvalue;
}