#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_absv(const unsigned int nSpheres, const float* velX, const float* velY, const float* velZ, float* d_absv) {
unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
if (my_sphere < nSpheres) {
float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
d_absv[my_sphere] = sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
}
}