#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void confidenceTransformKernel(const int width, const int height, const float threshold, const float gamma, const float clampedValue, const float* inputConfidence, float* outputConfidence) {
uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
if (x >= width || y >= height) return;
float inputValue = inputConfidence[y * width + x];
if (inputValue < threshold) {
outputConfidence[y * width + x] = 0;
} else {
outputConfidence[y * width + x] = powf(inputValue, gamma);
}
}