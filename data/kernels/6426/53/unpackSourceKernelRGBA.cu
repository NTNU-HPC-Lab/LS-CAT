#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpackSourceKernelRGBA(uint32_t* dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < width && y < height) {
// yeah, we could use a memcpy
uint32_t val;
surf2Dread(&val, src, x * sizeof(uint32_t), y);
dst[y * pitch + x] = val;
}
}