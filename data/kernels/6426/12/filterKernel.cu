#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void filterKernel(unsigned char* data, unsigned width, unsigned height, unsigned hStride, unsigned vStride, bool wrapAround) {
unsigned columnId = blockIdx.x * blockDim.x + threadIdx.x;

if (columnId < width) {
unsigned char* colp = data + columnId * hStride;
unsigned step = width * hStride * vStride;
{
uint32_t prev = colp[step];
// boundary condition
{
uint32_t pprev;
if (wrapAround) {
pprev = colp[step * (height - 1)];
} else {
pprev = prev;
}
uint32_t v = colp[0];
colp[0] = (2 * v + pprev + prev) >> 2;
}
__syncthreads();  // because of if
for (unsigned row = 2; row < height; row += 2) {
uint32_t next = colp[step * (row + 1)];
uint32_t v = colp[step * row];
colp[step * row] = (2 * v + next + prev) >> 2;
prev = next;
}
}
}
}