#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void onBothBufferOperatorKernel(const int warpWidth, const int input0OffsetX, const int input0OffsetY, const int input0Width, const int input0Height, const uint32_t* input0Buffer, const int input1OffsetX, const int input1OffsetY, const int input1Width, const int input1Height, const uint32_t* input1Buffer, const int outputOffsetX, const int outputOffsetY, const int outputWidth, const int outputHeight, uint32_t* outputMask) {
// calculate normalized texture coordinates
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < outputWidth && y < outputHeight) {
uint32_t v = 0;
const int outputX = x + outputOffsetX;
const int outputY = y + outputOffsetY;
const int input0X = (outputX + warpWidth - input0OffsetX) % warpWidth;
const int input0Y = (outputY - input0OffsetY);
const int input1X = (outputX + warpWidth - input1OffsetX) % warpWidth;
const int input1Y = (outputY - input1OffsetY);
if (input1X >= 0 && input1X < input1Width && input1Y >= 0 && input1Y < input1Height && input0X >= 0 &&
input0X < input0Width && input0Y >= 0 && input0Y < input0Height) {
if (input0Buffer[input0Y * input0Width + input0X] > 0 && input1Buffer[input1Y * input1Width + input1X] > 0) {
v = 1;
} else {
v = 0;
}
}
outputMask[y * outputWidth + x] = v;
}
}