#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpackSourceKernelGrayscale16(uint16_t* dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < width && y < height) {
float val;
surf2Dread(&val, src, x * sizeof(float), y);
const float inMilliMeters = val * 1000.f;
const uint16_t u16 = (uint16_t)max(0.f, min((float)USHRT_MAX, round(inMilliMeters)));
dst[y * pitch + x] = u16;
}
}