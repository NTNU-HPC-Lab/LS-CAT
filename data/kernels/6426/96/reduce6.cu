#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce6(const float* g_idata, float* g_odata, float* g_omask, unsigned int n) {
extern __shared__ float sharedData[];
float* sdata = &sharedData[0];
float* smask = &sharedData[blockDim.x];

// perform first level of reduction,
// reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
unsigned int gridSize = blockDim.x * 2 * gridDim.x;
sdata[tid] = 0;
smask[tid] = 0;

// we reduce multiple elements per thread.  The number is determined by the
// number of active thread blocks (via gridSize).  More blocks will result
// in a larger gridSize and therefore fewer elements per thread
while (i < n) {
sdata[tid] += g_idata[i];
smask[tid] += (g_idata[i] > 0 ? 1 : 0);
if (i + blockDim.x < n) {
sdata[tid] += g_idata[i + blockDim.x];
smask[tid] += (g_idata[i + blockDim.x] > 0 ? 1 : 0);
}
i += gridSize;
}
__syncthreads();

// do reduction in shared mem
if (blockDim.x >= 512) {
if (tid < 256) {
sdata[tid] += sdata[tid + 256];
smask[tid] += smask[tid + 256];
}
__syncthreads();
}
if (blockDim.x >= 256) {
if (tid < 128) {
sdata[tid] += sdata[tid + 128];
smask[tid] += smask[tid + 128];
}
__syncthreads();
}
if (blockDim.x >= 128) {
if (tid < 64) {
sdata[tid] += sdata[tid + 64];
smask[tid] += smask[tid + 64];
}
__syncthreads();
}

if (tid < 32) {
if (blockDim.x >= 64) {
sdata[tid] += sdata[tid + 32];
smask[tid] += smask[tid + 32];
}
if (blockDim.x >= 32) {
sdata[tid] += sdata[tid + 16];
smask[tid] += smask[tid + 16];
}
if (blockDim.x >= 16) {
sdata[tid] += sdata[tid + 8];
smask[tid] += smask[tid + 8];
}
if (blockDim.x >= 8) {
sdata[tid] += sdata[tid + 4];
smask[tid] += smask[tid + 4];
}
if (blockDim.x >= 4) {
sdata[tid] += sdata[tid + 2];
smask[tid] += smask[tid + 2];
}
if (blockDim.x >= 2) {
sdata[tid] += sdata[tid + 1];
smask[tid] += smask[tid + 1];
}
}

// write result for this block to global mem
if (tid == 0) {
g_odata[blockIdx.x] = sdata[0];
g_omask[blockIdx.x] = smask[0];
}
}