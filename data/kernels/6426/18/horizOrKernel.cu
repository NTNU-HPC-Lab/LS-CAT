#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void horizOrKernel(const uint32_t* __restrict__ contrib, uint32_t* __restrict__ rowHasImage, unsigned panoWidth, unsigned panoHeight) {
unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
const uint32_t* rowp = contrib + panoWidth * row;

if (row < panoHeight) {
uint32_t accum = 0;
for (unsigned col = 0; col < panoWidth; ++col) {
accum |= rowp[col];
}
rowHasImage[row] = accum;
}
}