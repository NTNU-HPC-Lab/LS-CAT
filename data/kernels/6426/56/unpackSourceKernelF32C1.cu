#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpackSourceKernelF32C1(float* dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < width && y < height) {
// yeah, we could use a memcpy
float val;
surf2Dread(&val, src, x * sizeof(float), y);
dst[y * pitch + x] = val;
}
}