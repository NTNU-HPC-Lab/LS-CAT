#include "hip/hip_runtime.h"
#include "includes.h"
#define LOG 0

/*
* An implementation of parallel reduction using nested kernel launches from
* CUDA kernels. This version adds optimizations on to the work in
* nestedReduce.cu.
*/

// Recursive Implementation of Interleaved Pair Approach
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check
if (idx >= n) return;

// in-place reduction in global memory
for (int stride = 1; stride < blockDim.x; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}