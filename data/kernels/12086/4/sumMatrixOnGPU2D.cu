#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int idx = iy * NX + ix;

if (ix < NX && iy < NY)
{
C[idx] = A[idx] + B[idx];
}
}