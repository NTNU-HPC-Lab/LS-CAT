#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON 		0.01f
#define L2HYS_EPSILONHYS	1.0f
#define L2HYS_CLIP			0.2f
#define data_h2y            30
//long h_windowx=Imagewidth/Windowx;
//long h_windowy=ImageHeight/Windowy;
//dim3 blocks(h_windowx,h_windowy);//h_windowx=ImageWidth/Windowx,h_windowy=ImageHeight/Windowy
//dim3 threads(Windowx,Windowy);//Ã¿Ò»¸öÏß³Ì¿é¼ÆËãÒ»¸öcellµÄÌØÕ÷Á¿

//dim3 block(18,7);//Ò»¸öcell·Ö18¸ö½Ç¶È·½Ïò,Ò»¸ö·½Ïò7¸öcell£¬
__global__ void normalizeL2Hys(float *in,float *out)
{
int bid=blockIdx.x;
int tid=threadIdx.x;
// Sum the vector
float sum = 0;

float *t_ftemp=in+bid*30;
float *t_foutemp=out+bid*30;
sum+=t_ftemp[tid]*t_ftemp[tid];
__syncthreads();
// Compute the normalization term
float norm = 1.0f/(rsqrt(sum) + L2HYS_EPSILONHYS * 30);
t_foutemp[tid]=t_ftemp[tid]*norm;
__syncthreads();


}