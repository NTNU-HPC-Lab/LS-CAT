#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myhistKernel(unsigned char * buffer,unsigned int * histo)
{
__shared__ unsigned int temp[256];

int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int offset = x + y * blockDim.x * gridDim.x;

temp[threadIdx.x]=0;
__syncthreads();

atomicAdd( &temp[buffer[offset]], 1 );

__syncthreads();
atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}