#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON 		0.01f
#define L2HYS_EPSILONHYS	1.0f
#define L2HYS_CLIP			0.2f
#define data_h2y            30
//long h_windowx=Imagewidth/Windowx;
//long h_windowy=ImageHeight/Windowy;
//dim3 blocks(h_windowx,h_windowy);//h_windowx=ImageWidth/Windowx,h_windowy=ImageHeight/Windowy
//dim3 threads(Windowx,Windowy);//Ã¿Ò»¸öÏß³Ì¿é¼ÆËãÒ»¸öcellµÄÌØÕ÷Á¿

//dim3 block(18,7);//Ò»¸öcell·Ö18¸ö½Ç¶È·½Ïò,Ò»¸ö·½Ïò7¸öcell£¬
__global__ void smooth(float *in,float *out)
{
int k,j,i;
int m_nBIN=10;
float *m_pCellFeatures=in;
int t_nLineWidth=70;
float t_pTemp[10];
for ( k = 0; k < 18; ++k )//18
{
for ( j = 0; j < 7; ++j )//7
{
for ( i = 0; i< 10; ++i )//10
{
int t_nLeft;
int t_nRight;
t_nLeft = ( i - 1 + m_nBIN ) % m_nBIN;
t_nRight = ( i + 1 ) % m_nBIN;

t_pTemp[i] = m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + i] * 0.8f
+ m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + t_nLeft] * 0.1f
+ m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + t_nRight] * 0.1f;
}

for ( i = 0; i < m_nBIN; ++i )
{
out[k * t_nLineWidth + j * m_nBIN + i] = t_pTemp[i];
}
}
}

}