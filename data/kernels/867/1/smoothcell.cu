#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON 		0.01f
#define L2HYS_EPSILONHYS	1.0f
#define L2HYS_CLIP			0.2f
#define data_h2y            30
//long h_windowx=Imagewidth/Windowx;
//long h_windowy=ImageHeight/Windowy;
//dim3 blocks(h_windowx,h_windowy);//h_windowx=ImageWidth/Windowx,h_windowy=ImageHeight/Windowy
//dim3 threads(Windowx,Windowy);//Ã¿Ò»¸öÏß³Ì¿é¼ÆËãÒ»¸öcellµÄÌØÕ÷Á¿

//dim3 block(18,7);//Ò»¸öcell·Ö18¸ö½Ç¶È·½Ïò,Ò»¸ö·½Ïò7¸öcell£¬
__global__ void smoothcell(float *in,float *out){
int t_nleft,t_nright;
t_nleft=(threadIdx.x-1+10)%10;
t_nright=(threadIdx.x+1)%10;
float *t_ptemp,t_ftemp[10];
t_ptemp=in+blockIdx.x*70+blockIdx.y*10;//+threadIdx.y)*0.8f+0.1f*(in+blockIdx.x*70+threadIdx.x*10+t_left)
/*__syncthreads();*/
if(t_ptemp)
t_ftemp[threadIdx.x]=t_ptemp[threadIdx.x]*0.8f+0.1f*t_ptemp[t_nleft]+0.1f*t_ptemp[t_nright];
__syncthreads();
out[blockIdx.x*70+blockIdx.y*10+threadIdx.x]=t_ftemp[threadIdx.x];
__syncthreads();
}