#include "hip/hip_runtime.h"
#include "includes.h"



#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);



//__global__ void markNonPrimeKernel(char* dev_chunk, number_type* min_primes, number_type currentValue, number_type currentValueSqr,
//	const number_type startValue, const number_type endValue, const int thread_size)
//{
//	const auto myThreadId = blockIdx.x * block_size + threadIdx.x;
//	const auto myStartValue = startValue + myThreadId * thread_size;
//	auto myEndValue = myStartValue + thread_size;
__global__ void getNextPrimeFast(char* dev_chunk, number_type currentValue, const number_type startValue, const number_type endValue, number_type* d_ans)
{
auto threadId = threadIdx.x;
if (threadId == 0)
{
for (auto i = currentValue + 1; i < endValue; i++)
{
auto number = dev_chunk[i];
if (number == 1)
{
*d_ans = i;
return;
}
}
}
}