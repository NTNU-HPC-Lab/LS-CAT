#include "hip/hip_runtime.h"
#include "includes.h"



#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);



//__global__ void markNonPrimeKernel(char* dev_chunk, number_type* min_primes, number_type currentValue, number_type currentValueSqr,
//	const number_type startValue, const number_type endValue, const int thread_size)
//{
//	const auto myThreadId = blockIdx.x * block_size + threadIdx.x;
//	const auto myStartValue = startValue + myThreadId * thread_size;
//	auto myEndValue = myStartValue + thread_size;
__global__ void markNonPrimeKernel(char* dev_chunk, number_type currentValue, number_type currentValueSqr, const number_type startValue, const number_type endValue, const int thread_size)
{
const auto myThreadId = blockIdx.x * block_size + threadIdx.x;
const auto myStartValue = startValue + myThreadId * thread_size;
auto myEndValue = myStartValue + thread_size;
if (myEndValue > endValue)
{
myEndValue = endValue;
}

number_type offset = 1;
// if current min first is set then we can offset by currentValue but if
// the number i is odd (which we can make sure of) then we can increment by
// currentValue * 2 as then we skip all even numbers in between which we dont need anyway
// as they will be already marked in case of 2
const int offsetMultiplier = (currentValue == 2) ? 1 : 2; //

auto updated_start = myStartValue;
if (updated_start != 0) // in case of zero first statement will underflow and will lead to max value
{
updated_start = myStartValue - myStartValue % currentValue;
if (updated_start % 2 == 0) // if even make it odd as only odd numbers can be marked off
//(even are done in case of 2, in which case subtracting 2 will still make it even)
{
updated_start -= currentValue;
}
}

if (updated_start < currentValueSqr)
updated_start = currentValueSqr;
offset = currentValue * offsetMultiplier;

for (auto i = updated_start; i < myEndValue; i += offset)
{
dev_chunk[i - startValue] = 0; // cancel that number, min is already marked, offset is current number
}
}