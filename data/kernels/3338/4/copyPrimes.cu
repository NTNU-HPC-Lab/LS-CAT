#include "hip/hip_runtime.h"
#include "includes.h"



#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);



//__global__ void markNonPrimeKernel(char* dev_chunk, number_type* min_primes, number_type currentValue, number_type currentValueSqr,
//	const number_type startValue, const number_type endValue, const int thread_size)
//{
//	const auto myThreadId = blockIdx.x * block_size + threadIdx.x;
//	const auto myStartValue = startValue + myThreadId * thread_size;
//	auto myEndValue = myStartValue + thread_size;
__global__ void copyPrimes(char* dev_chunk, number_type* base_index_arr, number_type* primes_arr, const number_type startValue, const number_type endValue, const int thread_size)
{
const auto my_thread_id = blockIdx.x * block_size + threadIdx.x;
auto my_start_value = startValue + my_thread_id * thread_size;
auto my_end_value = my_start_value + thread_size;
if (my_end_value > endValue)
{
my_end_value = endValue;
}
unsigned long index = base_index_arr[my_thread_id];

if (my_start_value == 0)
{
primes_arr[index] = 2; // add first prime 2, cause all others are odd
index++;
}
if (my_start_value % 2 == 0) // make odd, cause prime can only be in odd place
{
my_start_value += 1;
}

for (auto i = my_start_value; i < my_end_value; i+=2)
{
const auto current_status = dev_chunk[i - startValue];
if (current_status == 1)
{
primes_arr[index] = i;
index += 1;
}
}
}