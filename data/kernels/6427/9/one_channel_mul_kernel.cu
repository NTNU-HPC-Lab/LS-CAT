#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void one_channel_mul_kernel(const float *data_l, const float *data_r, float *result, int channel_total, int total)
{
int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
int one_ch_idx = idx  % (2 * channel_total);

if (idx / 2 < total) {
result[idx] = data_l[idx] * data_r[one_ch_idx] - data_l[idx + 1] * data_r[one_ch_idx + 1];
result[idx + 1] = data_l[idx] * data_r[one_ch_idx + 1] + data_l[idx + 1] * data_r[one_ch_idx];
}
}