#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void sqr_norm_kernel(const float *in, float *block_res, int total)
{
extern __shared__ float sdata[];
int in_idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
int i = threadIdx.x;
unsigned ins = blockDim.x;

if (in_idx >= total * 2)
sdata[i] = 0;
else
sdata[i] = in[in_idx] * in[in_idx] + in[in_idx + 1] * in[in_idx + 1];

for (unsigned outs = (ins + 1) / 2; ins > 1; ins = outs, outs = (outs + 1) / 2) {
__syncthreads();
if (i + outs < ins)
sdata[i] += sdata[i + outs];
}

if (i == 0)
block_res[blockIdx.x] = sdata[0];
}