#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
if (crossTimeIndex < numSims) {
hiprandState_t state;
hiprand_init (blockIdx.x * 1000 + threadIdx.x + seconds, 0, 0, &state);
double random;
int start = (threadIdx.x + blockIdx.x * blockDim.x) * N;

crossTimes[crossTimeIndex] = 0;
results[start] = 0.0;
bool crossed = false;

for (int j = start + 1; j < start + N; j++) {
random = hiprand_normal_double(&state);
results[j] = results[j-1] + random * sqrt((double) T / N);
if (!crossed && results[j] >= upperThreshold) {
crossTimes[crossTimeIndex] = j - start;
crossed = true;
}
else if (!crossed && results[j] <= lowerThreshold) {
crossTimes[crossTimeIndex] = -1 * (j - start);
crossed = true;
}
}

if (!crossed) {
atomicAdd(dev_failCross, 1);
}

}


/*
Generate 2 doubles at once. Test later to see if this is more efficient:
double hiprand_normal2_double (state);
*/

}