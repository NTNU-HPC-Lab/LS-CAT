#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double upperThreshold, double deviceID) {

// a variable to keep track of this simulation's position in the crossTimes array
int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

if (crossTimeIndex < numSims) {

// create random number generator
hiprandState_t state;
hiprand_init (blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);
double random;

// starting position of this siulation in results array
int start = (threadIdx.x + blockIdx.x * blockDim.x) * N;

// set default value of cross time for this simulation to 0, since the simulation hasn't crossed the threshold yet
crossTimes[crossTimeIndex] = 0;

// starting point of path is 0
results[start] = 0.0;

// boolean to keep track of whether this path has crossed
bool crossed = false;

for (int j = start + 1; j < start + N; j++) {
// generate random number
random = hiprand_normal_double(&state);

//calculate next step of path
results[j] = results[j-1] + random * sqrt((double) T / N);

// store crossing time as positive value if it has crossed the upper threshold. Negative value if crossed the lower threshold
if (!crossed && results[j] >= upperThreshold) {
crossTimes[crossTimeIndex] = j - start;
crossed = true;
}
}

}

}