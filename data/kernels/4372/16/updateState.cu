#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateState(double *B, double *external, double *lamBeta, int dim, float timestep, double noise, int length, int totalIterations, int iterationNum) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
if (index >= length && index < length + dim) {
int neuronNum = index % dim;
double lam = lamBeta[neuronNum * 2];
double beta = lamBeta[neuronNum * 2 + 1];

double input = B[index] + external[neuronNum * (totalIterations) + iterationNum];
double old_output = B[index - dim];
double d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * lam * (input - beta)));

// create random number generator
hiprandState_t state;
hiprand_init (blockIdx.x * 1000 + threadIdx.x + clock64(), 0, 0, &state);
float random = hiprand_normal(&state);
double guassian_noise = noise * random * sqrt(timestep);
B[index] = old_output + d_layers * timestep + guassian_noise;
}
}