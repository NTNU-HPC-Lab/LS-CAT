#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateState(float *B, float *external, int dim, float timestep, float noise, int length, int totalIterations, int iterationNum, float L, float M) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
if (index >= length && index < length + dim) {
int neuronNum = index % dim;
float input = B[index] + external[neuronNum * (totalIterations) + iterationNum];
float old_output = B[index - dim];
float d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * L * (input - M)));

// create random number generator
hiprandState_t state;
hiprand_init (blockIdx.x * 1000 + threadIdx.x + clock64(), 0, 0, &state);
float random = hiprand_normal(&state);
float guassian_noise = noise * random * sqrt(timestep);
B[index] = old_output + d_layers * timestep + guassian_noise;
}
}