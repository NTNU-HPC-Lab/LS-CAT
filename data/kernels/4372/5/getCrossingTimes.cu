#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getCrossingTimes(double *results, int *crossTimes, int N, int numSims, int lowerThreshold, int upperThreshold) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < N * numSims) {
if (crossTimes[tid/N] == 0) {
if (results[tid] <= lowerThreshold) {
crossTimes[tid/N] = tid % N;
}
else if (results[tid] >= upperThreshold) {
crossTimes[tid/N] = tid % N;
}
}
tid += blockDim.x + gridDim.x;
}
}