#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float3 addCuda(float3 a, float3 b) {
return{ a.x + b.x, a.y + b.y, a.z + b.z };
}
__device__ inline float3 multiplyCuda(float a, float3 b) {
return{ a * b.x, a * b.y, a * b.z };
}
__device__ inline float euclideanLenCuda(float3 a, float3 b, float d) {
float mod = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) + (b.z - a.z) * (b.z - a.z);
return expf(-mod / (2.0f * d * d));
}
__global__ void bilateralFilterCudaKernel(float3 * dev_input, float3 * dev_output, float l2norm, int width, int height, int radius)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x<width) && (y<height))
{
float sum = 0.0f;
float3 t = { 0.f, 0.f, 0.f };
float3 center = dev_input[y * width + x];
int r = radius;

float domainDist = 0.0f, colorDist = 0.0f, factor = 0.0f;

for (int i = -r; i <= r; i++) {
int crtY = y + i;
if (crtY < 0)				crtY = 0;
else if (crtY >= height)   	crtY = height - 1;

for (int j = -r; j <= r; ++j) {
int crtX = x + j;
if (crtX < 0) 				crtX = 0;
else if (crtX >= width)	 	crtX = width - 1;

float3 curPix = dev_input[crtY * width + crtX];
domainDist = c_gaussian[r + i] * c_gaussian[r + j];
colorDist = euclideanLenCuda(curPix, center, l2norm);
factor = domainDist * colorDist;
sum += factor;
t = addCuda(t, multiplyCuda(factor, curPix));
}
}

dev_output[y * width + x] = multiplyCuda(1.f / sum, t);
}
}