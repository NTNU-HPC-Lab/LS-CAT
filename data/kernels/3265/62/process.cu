#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float dothings(int t,int sz, float *input){
float ans = 0;
for(int i=0;i<12;++i){
ans += input[(i+t)%sz];
}
return ans;
}
__global__ void process(int N_step, int N_inst, float *input, float *output){
int g_id = blockIdx.x * blockDim.x + threadIdx.x;
if(g_id >= N_inst) return;
float local_data[VEC_SIZE];
float ans = 0.;
for(int i=0;i<VEC_SIZE;++i) local_data[i] = input[VEC_SIZE * g_id + i];
for(int t=0;t<N_step;++t){
ans += dothings(t, VEC_SIZE, local_data);
}
output[g_id] = ans;
return;
}