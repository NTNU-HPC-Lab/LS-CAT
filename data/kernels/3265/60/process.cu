#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process(int N_step, int N_inst, float *input, float *output){
int g_id = blockIdx.x * blockDim.x + threadIdx.x;
if(g_id >= N_inst) return;
float ans = 0.;
for(int t=0;t<N_step;++t){
for(int i=0;i<12;++i){
ans += input[(i+t)%VEC_SIZE + VEC_SIZE * g_id];
}
}
output[g_id] = ans;
return;
}