#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sharedSum(int N, float *input, float *output){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i >= N) return;
__shared__ float tmp[BLOCK_SIZE];
memset(tmp, 0, sizeof(tmp));
float a = input[i];
for(int j=0;j<BLOCK_SIZE;++j){
atomicAdd(tmp + j, a);
}
__syncthreads();
output[blockDim.x*blockIdx.x + threadIdx.x] = tmp[threadIdx.x];
return;
}