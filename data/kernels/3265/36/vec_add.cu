#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vec_add(int N, int *A, int *B, int *C){
int i = threadIdx.x + blockIdx.x * blockDim.x;
if(i < N) C[0] = A[i] * B[i];
}