#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmulKernel(float *A, float *B, float *C, int rA, int cA, int cB){
int i = blockIdx.y*gridDim.x + blockIdx.x, j = threadIdx.y*blockDim.x + threadIdx.x;
if(i < rA && j < cB){
C[i*cB + j] = 0.;
for(int k=0;k<cA;++k) C[i*cB + j] += A[i*cA + k] * B[k*cB + j];
}
return;
}