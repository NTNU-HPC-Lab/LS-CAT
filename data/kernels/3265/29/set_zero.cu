#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_zero(double *A, double *I, int n, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n){
if (x != i){
if (y == i){
A[x*n + y] = 0;
}
}
}
}