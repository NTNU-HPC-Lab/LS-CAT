#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naiveKernel(int N, double *input, double *output){
int global_i = blockIdx.x * blockDim.x + threadIdx.x;
if(global_i < N){
for(int i=0;i<N;++i) output[global_i] += input[i];
output[global_i] /= N;
}
return ;
}