#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "simKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int N_stgy = 1;
int N_batch = 2;
double *alpha = NULL;
hipMalloc(&alpha, XSIZE*YSIZE);
double *mid = NULL;
hipMalloc(&mid, XSIZE*YSIZE);
double *gap = NULL;
hipMalloc(&gap, XSIZE*YSIZE);
int *late = NULL;
hipMalloc(&late, XSIZE*YSIZE);
int *pos = NULL;
hipMalloc(&pos, XSIZE*YSIZE);
int *rest_lag = NULL;
hipMalloc(&rest_lag, XSIZE*YSIZE);
double *prof = NULL;
hipMalloc(&prof, XSIZE*YSIZE);
double *last_prc = NULL;
hipMalloc(&last_prc, XSIZE*YSIZE);
int *cnt = NULL;
hipMalloc(&cnt, XSIZE*YSIZE);
double fee = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
simKernel<<<gridBlock,threadBlock>>>(N_stgy,N_batch,alpha,mid,gap,late,pos,rest_lag,prof,last_prc,cnt,fee);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
simKernel<<<gridBlock,threadBlock>>>(N_stgy,N_batch,alpha,mid,gap,late,pos,rest_lag,prof,last_prc,cnt,fee);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
simKernel<<<gridBlock,threadBlock>>>(N_stgy,N_batch,alpha,mid,gap,late,pos,rest_lag,prof,last_prc,cnt,fee);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}