#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smemKernel(int N, float *input, float *output){
int b_size = blockDim.x, b_idx = blockIdx.x, t_idx = threadIdx.x;
int global_i = b_size * b_idx + t_idx, n_chk = (N + SHARE_SIZE - 1)/SHARE_SIZE;
__shared__ float buff[SHARE_SIZE];
for(int q=0;q<n_chk;++q){
int left = q*SHARE_SIZE, right = min(left + SHARE_SIZE, N);
for(int i = t_idx + left; i < right; i += b_size) buff[i-left] = input[i];
__syncthreads();
if(global_i < N){
for(int i = left; i < right; ++i) output[global_i] += buff[i-left];
}
__syncthreads();
}
output[global_i] /= N;
return ;
}