#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process(int N_step, int N_inst, float *input, float *output){
int b_id = blockIdx.x, t_id = threadIdx.x;
if(b_id >= N_inst) return;
__shared__ float ans;
float val;
if(!t_id) ans = 0;
if(t_id < VEC_SIZE) val = input[VEC_SIZE * b_id + t_id];
__syncthreads();
for(int t=0;t<N_step;++t){
int start = t%VEC_SIZE;
if(t_id >= start && t_id < min(start + 12, VEC_SIZE)) atomicAdd(&ans, val);
if(start + 12 > VEC_SIZE && t_id < start + 12 - VEC_SIZE) atomicAdd(&ans, val);
__syncthreads();
}
output[b_id] = ans;
return;
}