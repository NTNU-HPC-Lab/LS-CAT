#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaGetShiftedMidPrice(int N_inst, int batch_size, float *alphas, float *mid, float *shifted_prc){
int b_sz = blockDim.x, b_id = blockIdx.x, t_id = threadIdx.x;
if(b_id < N_inst){
for(int i=t_id; i<batch_size; i += b_sz){
shifted_prc[b_id * batch_size + i] = (1. + alphas[b_id * batch_size + i]) * mid[i];
}
}
}