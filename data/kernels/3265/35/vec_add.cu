#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vec_add(int N, int *A, int *B, int *C){
int i = threadIdx.x + blockIdx.x * blockDim.x;
// assert( i<N );
if(i < N) C[i] = A[i] + B[i];
}