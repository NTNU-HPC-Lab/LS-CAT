#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaGetError(int N, double *ana, double *cur, double *e_sum){
// Parallelly compute the error
int index = blockIdx.x*blockDim.x + threadIdx.x;
if(index < (N+1)*(N+1)) (*e_sum) += (ana[index] - cur[index])*(ana[index] - cur[index]);
return;
}