#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatterSum(int N, float *input, float *output){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i >= N) return;
float a = input[i];
for(int j=0;j<N;++j){
atomicAdd(output+(j+i)%N, a);
}
return;
}