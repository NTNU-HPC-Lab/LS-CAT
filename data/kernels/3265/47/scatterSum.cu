#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatterSum(int N, float *input, float *output){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i >= N) return;
for(int j=0;j<N;++j){
atomicAdd(output+j, input[i]);
// if(i<N/2) atomicAdd(output+j, input[i]);
// atomicAdd(output+j, i<N/2: input[i]: 0.);
}
return;
}