#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simKernel(int N_stgy, int N_batch, float *alpha, float *mid, float *gap, int *late, int *pos, int *rest_lag, float *prof, float *last_prc, int *cnt, float fee){
int global_i = blockIdx.x*blockDim.x + threadIdx.x;
if( global_i >= N_stgy) return;
int start = global_i*N_batch + rest_lag[global_i], end = global_i*N_batch + N_batch, i;
for(i = start; i<end; ++i) if(alpha[i]*mid[i%N_batch]>gap[i%N_batch] + fee || alpha[i]*mid[i%N_batch]<-gap[i%N_batch] - fee){
if(alpha[i]*mid[i%N_batch]>gap[i%N_batch]+fee && pos[global_i]<1){
last_prc[global_i] = mid[i%N_batch] + gap[i%N_batch] + fee;
prof[global_i] -= (1-pos[global_i])*last_prc[global_i];
cnt[global_i] += 1-pos[global_i];
pos[global_i] = 1;
i += late[i%N_batch];
}
else if(alpha[i]*mid[i%N_batch]<-gap[i%N_batch]-fee && pos[global_i]>-1){
last_prc[global_i] = mid[i%N_batch] - gap[i%N_batch] - fee;
prof[global_i] += (pos[global_i]+1)*last_prc[global_i];
cnt[global_i] += pos[global_i]+1;
pos[global_i] = -1;
i += late[i%N_batch];
}
}
rest_lag[global_i] = i-end;
}