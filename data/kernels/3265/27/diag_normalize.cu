#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diag_normalize(double *A, double *I, int n, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n)
if (x == y && x == i){
I[x*n + y] /= A[i*n + i];
A[x*n + y] /= A[i*n + i];
}

}