#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gatherSum(int N, float *input, float *output){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i >= N) return;
for(int j=0;j<N;++j){
output[i] += input[j];
}
return;
}