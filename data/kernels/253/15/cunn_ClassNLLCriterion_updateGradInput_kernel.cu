#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget) {
register int i, j, t;
for (i = threadIdx.x; i < nframe; i += NTHREADS) {
for (j = 0; j < ntarget; ++j) {
t = (int)target[i * ntarget + j] - 1;
if (t >= 0)
gradInput[i * ndim + t] = grad;
}
}
}