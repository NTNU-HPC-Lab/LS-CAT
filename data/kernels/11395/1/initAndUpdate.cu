#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initAndUpdate( float *D_oldVal, float *D_currVal, int tpoints, int nsteps )
{
int j = blockDim.x * blockIdx.x + threadIdx.x;
if ( j < tpoints )
{
j += 1;
/* Calculate initial values based on sine curve */
/* Initialize old values array */
float x = ( float )( j - 1 ) / ( tpoints - 1 );
D_oldVal[j] = D_currVal[j] = sin ( 6.2831853f * x );
int i;
/* global endpoints */
if ( ( j == 1 ) || ( j  == tpoints ) )
{
D_currVal[j] = 0.0;
}
else
{
/* Update values for each time step */
for ( i = 1; i <= nsteps; i++ )
{
/* Update old values with new values */
float newVal = ( 2.0 * D_currVal[j] ) - D_oldVal[j] + ( 0.09f * ( -2.0 ) * D_currVal[j] );
D_oldVal[j] = D_currVal[j];
D_currVal[j] = newVal;
}
}
}
}