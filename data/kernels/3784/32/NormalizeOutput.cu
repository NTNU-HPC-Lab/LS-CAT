#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_elements; idx += blockDim.x * gridDim.x) {
to_normalize[idx * 3] = batch_index;
to_normalize[idx * 3 + 1] = class_index;
to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
}
}