#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _roll_array( const float* array, const long* step, float* new_array, const int b, const int n, const int d ) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index >= b * n * d)
return;

const int c_b = index / (n * d);
const int c_n = (index - c_b * n * d) / d;
const int c_d = index % d;

const float c_array_element = array[c_b * n * d + c_n * d + c_d];
float* c_new_array = &new_array[c_b * n * d];

int c_step = int(step[c_b]);
int new_n = ((c_n + c_step) % n + n) % n;
int position = new_n * d + c_d;

c_new_array[position] = c_array_element;
}