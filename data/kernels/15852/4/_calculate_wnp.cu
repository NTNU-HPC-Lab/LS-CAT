#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _calculate_wnp( const long* edge_num, const long* edge_start_idx, float* weight, long* ind, const int b, const int n, const int orig_p_num, const int p_num ) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index >= b * n * orig_p_num)
return;

const int c_b = index / (n * orig_p_num);
const int c_n = (index - c_b * n * orig_p_num) / orig_p_num;
const int c_edge_idx = index % orig_p_num;

const long c_edge_num = edge_num[index];
const int c_start_idx = int(edge_start_idx[index]);
float* c_weight = &weight[c_b * n * p_num + c_n * p_num + c_start_idx];
long* c_ind = &ind[c_b * n * p_num * 2 + c_n * p_num * 2 + c_start_idx * 2];

for (long i = 0; i < c_edge_num; i++) {
c_weight[i] = float(i) / float(c_edge_num);
c_ind[i * 2] = long(c_edge_idx);
c_ind[i * 2 + 1] = long((c_edge_idx + 1) % orig_p_num);
}
}