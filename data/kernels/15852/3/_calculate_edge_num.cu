#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _calculate_edge_num( long* edge_num, const long* edge_num_sum, const long* edge_idx_sort, const int b, const int n, const int orig_p_num, const long p_num ) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index >= b * n)
return;

const int c_b = index / n;
const int c_n = index % n;

long* c_edge_num = &edge_num[c_b * n * orig_p_num + c_n * orig_p_num];
const long c_edge_num_sum = edge_num_sum[c_b * n + c_n];
const long* c_edge_idx_sort = &edge_idx_sort[c_b * n * orig_p_num + c_n * orig_p_num];

if (c_edge_num_sum == p_num)
return;

if (c_edge_num_sum < p_num)
c_edge_num[c_edge_idx_sort[0]] += p_num - c_edge_num_sum;
else {
int id = 0;
long pass_num = c_edge_num_sum - p_num;
while (pass_num > 0) {
long edge_idx = c_edge_idx_sort[id];
if (c_edge_num[edge_idx] > pass_num) {
c_edge_num[edge_idx] -= pass_num;
pass_num = 0;
} else {
pass_num -= c_edge_num[edge_idx] - 1;
c_edge_num[edge_idx] = 1;
id += 1;
}
}
}
}