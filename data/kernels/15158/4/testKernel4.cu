#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel4(float *data1, float *data2)
{
float t = 0.0f;
float c = 0.0f;

//printf("d = %f\n", data1[NX*blockIdx.x + threadIdx.x]);

if(blockIdx.x > 0)
{
t += (data1[NX*(blockIdx.x-1)+threadIdx.x] - data1[NX*blockIdx.x + threadIdx.x]);
c += 1.0f;
}
if(blockIdx.x < NX-1)
{
t += (data1[NX*(blockIdx.x+1)+threadIdx.x] - data1[NX*blockIdx.x+threadIdx.x]);
c+=1.0f;
}
if(threadIdx.x > 0)
{
t += (data1[NX*blockIdx.x+threadIdx.x-1] - data1[NX*blockIdx.x+threadIdx.x]);
c+=1.0f;
}
if(threadIdx.x < NX-1)
{
t += (data1[NX*blockIdx.x+threadIdx.x+1] - data1[NX*blockIdx.x+threadIdx.x]);
c+=1.0f;
}
//printf("block %i, %i, %i\n", blockIdx.x, threadIdx.x, 1024*blockIdx.x+threadIdx.x);
//data2[1024*blockIdx.x+threadIdx.x] = 2*data1[1024*blockIdx.x+threadIdx.x];
if(blockIdx.x == 0)
data2[NX*blockIdx.x+threadIdx.x] = 1.0;
else
data2[NX*blockIdx.x+threadIdx.x] = data1[NX*blockIdx.x+threadIdx.x] + t/c*DIFF_RATE;
return;
}