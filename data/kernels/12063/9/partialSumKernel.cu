#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void partialSumKernel(int *X, int N)
{
__shared__ int partialSum[BLOCK_SIZE];
int tx = threadIdx.x;
int i = blockIdx.x * blockDim.x + tx;

if (i < N) {
partialSum[tx] = X[i];
partialSum[tx + blockDim.x] = X[i + gridDim.x * blockDim.x];
//printf("X[%d + %d * %d] = %d\n", i,gridDim.x, blockDim.x, X[i + gridDim.x * blockDim.x]);
}
else
partialSum[tx] = 0; // last block may pad with 0's

for (int stride = blockDim.x; stride > 0; stride = stride/2)
{
__syncthreads();
if (tx < stride) {
//printf("tx[%d], bx[%d]: %d + %d\n", tx, blockIdx.x, partialSum[tx], partialSum[tx + stride]);
partialSum[tx] += partialSum[tx + stride];
}
}
if (tx == 0)
X[blockIdx.x] = partialSum[tx];
}