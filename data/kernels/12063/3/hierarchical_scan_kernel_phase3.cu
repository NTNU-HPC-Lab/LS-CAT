#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hierarchical_scan_kernel_phase3(int *S, int *Y) {

int tx = threadIdx.x, bx = blockIdx.x;
int i = bx * SECTION_SIZE + tx;
//printf("Y[%d] = %.2f\n", i, Y[i]);

if (bx > 0)
{
for (int j = 0; j < SECTION_SIZE ; j += BLOCK_DIM ) {
Y[i + j] += S[bx - 1];
}
}
}