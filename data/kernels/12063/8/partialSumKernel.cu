#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void partialSumKernel(int *X, int N)
{
__shared__ int partialSum[2 * BLOCK_SIZE];
int tx = threadIdx.x;
int i = blockIdx.x * blockDim.x + tx;
partialSum[tx] = (i < N) ?  X[i] : 0;
partialSum[tx + blockDim.x] = 0;

for (int stride = blockDim.x; stride > 0; stride = stride/2)
{
__syncthreads();
if (tx <= stride) {
partialSum[tx] += partialSum[tx + stride];
//printf("tx[%d], bx[%d]: %d + %d\n", tx, blockIdx.x, partialSum[tx], partialSum[tx + stride]);
}
}
if (tx == 0)
X[blockIdx.x] = partialSum[tx];
}