#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize)
{
__shared__ float XY[SECTION_SIZE];
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < InputSize) {
XY[threadIdx.x] = X[i];
}

// Perform iterative scan on XY
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
float in;
__syncthreads();
if (threadIdx.x >= stride){
in = XY[threadIdx.x - stride];
}
__syncthreads();
if (threadIdx.x >= stride){
XY[threadIdx.x] += in;
}
}

__syncthreads();
Y[i] = XY[threadIdx.x];
}