#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize)
{
__shared__ float XY[SECTION_SIZE];
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < InputSize && threadIdx.x != 0) {
XY[threadIdx.x] = X[i - 1];
}
else {
XY[threadIdx.x] = 0;
}

if (threadIdx.x < InputSize)
{
// Perform iterative exclusive scan on XY
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
if (threadIdx.x >= stride) {
__syncthreads();
XY[threadIdx.x] += XY[threadIdx.x - stride];
}
}
Y[i] = XY[threadIdx.x];
}
}