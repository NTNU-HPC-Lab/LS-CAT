#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hierarchical_scan_kernel_phase1(int *X, int *Y, int *S) {
__shared__ int XY[SECTION_SIZE];
__shared__ int AUS[BLOCK_DIM];
int tx = threadIdx.x, bx = blockIdx.x;
int i = bx * SECTION_SIZE + tx;

if (i < INPUT_SIZE) {

// collaborative load in a coalesced manner
for (int j = 0; j < SECTION_SIZE; j+=BLOCK_DIM) {
XY[tx + j] = X[i + j];
}
__syncthreads();


// PHASE 1: scan inner own subsection
// At the end of this phase the last element of each subsection contains the sum of all alements in own subsection
for (int j = 1; j < SUBSECTION_SIZE; j++) {
XY[tx * (SUBSECTION_SIZE) + j] += XY[tx * (SUBSECTION_SIZE)+j - 1];
}
__syncthreads();


// PHASE 2: perform iterative kogge_stone_scan of the last elements of each subsections of XY loaded first in AUS
AUS[tx] = XY[tx * (SUBSECTION_SIZE)+(SUBSECTION_SIZE)-1];
int in;
for (unsigned int stride = 1; stride < BLOCK_DIM; stride *= 2) {
__syncthreads();
if (tx >= stride) {
in = AUS[tx - stride];
}
__syncthreads();
if (tx >= stride) {
AUS[tx] += in;
}
}
__syncthreads();

// PHASE 3: each thread adds to its elements the new value of the last element of its predecessor's section
if (tx > 0) {
for (unsigned int stride = 0; stride < (SUBSECTION_SIZE); stride++) {
XY[tx * (SUBSECTION_SIZE)+stride] += AUS[tx - 1];  // <--
}
}
__syncthreads();

// store the result into output vector
for (int j = 0; j < SECTION_SIZE; j += BLOCK_DIM) {
Y[i + j] = XY[tx + j];
}

//The last thread in the block writes the output value of the last element in the scan block to the blockIdx.x position of S
if (tx == BLOCK_DIM - 1) {
S[bx] = XY[SECTION_SIZE - 1];
}
}
}