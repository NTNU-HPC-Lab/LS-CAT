#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void efficient_Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize) {
__shared__ float XY[SECTION_SIZE];
__shared__ float AUS[BLOCK_DIM];
//int i = blockIdx.x * blockDim.x + threadIdx.x;

// Keep mind: Partition the input into blockDim.x subsections: i.e. for 8 threads --> 8 subsections

// collaborative load in a coalesced manner
for (int j = 0; j < SECTION_SIZE; j += blockDim.x) {
XY[threadIdx.x + j] = X[threadIdx.x + j];
}
__syncthreads();


// PHASE 1: scan inner own subsection
// At the end of this phase the last element of each subsection contains the sum of all alements in own subsection
for (int j = 1; j < SUBSECTION_SIZE; j++) {
XY[threadIdx.x * (SUBSECTION_SIZE)+j] += XY[threadIdx.x * (SUBSECTION_SIZE)+j - 1];
}
__syncthreads();


// PHASE 2: perform iterative kogge_stone_scan of the last elements of each subsections of XY loaded first in AUS
AUS[threadIdx.x] = XY[threadIdx.x * (SUBSECTION_SIZE)+(SUBSECTION_SIZE)-1];
float in;
for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
__syncthreads();
if (threadIdx.x >= stride) {
in = AUS[threadIdx.x - stride];
}
__syncthreads();
if (threadIdx.x >= stride) {
AUS[threadIdx.x] += in;
}
}
__syncthreads();


// PHASE 3: each thread adds to its elements the new value of the last element of its predecessor's section
if (threadIdx.x > 0) {
for (unsigned int stride = 0; stride < (SUBSECTION_SIZE); stride++) {
XY[threadIdx.x * (SUBSECTION_SIZE)+stride] += AUS[threadIdx.x - 1];  // <--
}
}
__syncthreads();


// store the result into output vector
for (int j = 0; j < SECTION_SIZE; j += blockDim.x) {
Y[threadIdx.x + j] = XY[threadIdx.x + j];
}
}