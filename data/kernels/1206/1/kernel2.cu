#include "hip/hip_runtime.h"
#include "includes.h"
/*
Problem 1: initialize array of size 32 to 0
Problem 2: change array size to 1024
Problem 3: create another kernel that adds i to array[ i ]
Problem 4: change array size 8000 (check answer to Problem 3 still works)
*/

//initialize array to 0

//add i to array[ i ]

__global__ void kernel2( int N, int *d_array ){
for( int i = 0; i < N; i++ ){
d_array[ i ] = i;
}
}