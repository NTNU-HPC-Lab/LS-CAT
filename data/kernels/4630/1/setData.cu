#include "hip/hip_runtime.h"
#include "includes.h"

/* https://zxi.mytechroad.com/blog/dynamic-programming/leetcode-730-count-different-palindromic-subsequences/ */


long kMod = 1000000007;

__global__ void setData(int *dp, int n) {
for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
dp[i * n + i] = 1;
}
}