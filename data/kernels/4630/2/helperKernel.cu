#include "hip/hip_runtime.h"
#include "includes.h"

/* https://zxi.mytechroad.com/blog/dynamic-programming/leetcode-730-count-different-palindromic-subsequences/ */


long kMod = 1000000007;

__global__ void helperKernel(char *S, int *dp, int n, long kMod, int len) {
for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n - len; i += blockDim.x * gridDim.x) {
int j = i + len; // jth element is the end of current string
if(S[i] == S[j]) { // if front and rear are the same
dp[i * n + j] = dp[(i + 1) * n + (j - 1)] * 2;
int left = i + 1;
int right = j - 1;

while(left <= right && S[left] != S[i]) {
left++;
}
while(left <= right && S[right] != S[i]) {
right--;
}

if(left == right) {
dp[i * n + j] += 1;
} else if(left > right) {
dp[i * n + j] += 2;
} else {
dp[i * n + j] -= dp[(left + 1) * n + (right - 1)];
}
} else {
dp[i * n + j] = dp[i * n + (j - 1)] + dp[((i + 1) * n) + j] - dp[(i + 1) * n + (j - 1)];
}

dp[i * n + j] = (dp[i * n + j] + kMod) % kMod; // perform positive modulo
}
//__syncthreads();
}