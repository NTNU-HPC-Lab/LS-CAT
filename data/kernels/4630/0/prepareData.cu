#include "hip/hip_runtime.h"
#include "includes.h"

/* https://zxi.mytechroad.com/blog/dynamic-programming/leetcode-730-count-different-palindromic-subsequences/ */


long kMod = 1000000007;

__global__ void prepareData(int *dp, int n) {
for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
for(int j = blockIdx.y * blockDim.y + threadIdx.y; j < n; j+= blockDim.y * gridDim.y) {
dp[i * n + j] = 0;
}
}
}