#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition ( const int *dev_a , const int *dev_b , int *dev_c)
{
//Get the id of thread within a block
unsigned short tid = blockDim.x*blockIdx.x+threadIdx.x;

if ( tid < N ) // check the boundry condition for the threads
dev_c [tid] = dev_a[tid] + dev_b[tid] ;

}