#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition (  int *dev_a ,  int *dev_b , int *dev_c)
{
//Lay ra id cua thread trong 1 block.
int tid = blockIdx.x ; // blockDim.x*blockIdx.x+threadIdx.x

if ( tid < N )
*(dev_c+tid) = *(dev_a+tid) + *(dev_b+tid) ;

}