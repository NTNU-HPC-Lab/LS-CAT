#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixAdd_CUDA(int *A, int *B, int *C) {
int i= blockIdx.y*blockDim.y+ threadIdx.y;
int j = blockIdx.x*blockDim.x+ threadIdx.x;
*(C + i*N + j) =  *(A + i*N + j)+ *(B + i*N + j);

}