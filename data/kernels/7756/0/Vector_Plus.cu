#include "hip/hip_runtime.h"
#include "includes.h"

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ))

__global__ void Vector_Plus ( int *AG ,  int *BG , int *CG)
{
int id = blockDim.x*blockIdx.x+threadIdx.x ;
if ( id < N )
*(CG+id)=*(AG+id)+ *(BG+id);

}