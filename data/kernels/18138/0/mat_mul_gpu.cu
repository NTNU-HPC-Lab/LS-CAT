#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mul_gpu(float* vec_one, float* vec_two, float* ret_vec, int vec_one_row, int vec_one_col, int vec_two_col) {
// compute global thread coordinates
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

// linearize coordinates for data access
int offset = row * vec_two_col + col;
// vec_one_col is equal to vec_two_row

if ((row < vec_one_row) && (col < vec_two_col)) {
float cum_sum = 0.0;
for (int k = 0; k < vec_one_col; k++) {
cum_sum += vec_one[row * vec_one_col + k] * vec_two[k * vec_two_col + col];
}
ret_vec[offset] = cum_sum;
}
}