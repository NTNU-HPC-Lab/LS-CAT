#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_calculation(float c0r, float c0i, float float_step, float imag_step, int *results, unsigned n, int W, int H, int inicial){

// index = m*x + y
const long unsigned globalIndex = blockDim.x*blockIdx.x + threadIdx.x;

// printf("%d %d\n", blockIdx.x, threadIdx.x);

if (globalIndex < n) {
//calcular os complexos na mão
int x = (globalIndex + inicial)/W;
int y = (globalIndex + inicial)%H;
// printf("%d %d    %d\n", x, y, n);
float point_r = c0r+x*float_step;
float point_i = c0i+y*imag_step;

// printf("%f %f\n", point_r, point_i);
const int M = 1000;

// valor Zj que falhou
// -1 se não tiver falhado
int j = -1;

//Valor da iteração passada
float old_r = 0;
float old_i = 0;
float aux = 0;

//Calcula o mandebrot
for(int i = 1; i <= M; i++){

//Calculo da nova iteração na mão
aux = (old_r * old_r) - (old_i * old_i) + point_r;
old_i = (2 * old_r * old_i) + point_i;
old_r = aux;

//abs(complex) = sqrt(a*a + b*b)
//Passei a raiz do abs para outro lado
if( ((old_r * old_r + old_i * old_i) > 4 )){
j = i;
break;
}
}
// printf("%d\n", j);
// printf("%d\n", j);

results[globalIndex] = j;
// printf("%d\n", j);
}
// else printf("oh boy\n");

}