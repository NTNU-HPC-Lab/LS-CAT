#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for(int i = tid; i < N; i+=stride){
c[i] = 2 * a[i] + b[i];
}
}