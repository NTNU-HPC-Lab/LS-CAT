#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThanEqScalar(float* mat, float val, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] <= val;
}