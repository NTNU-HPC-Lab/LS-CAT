#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches(float* images, float* patches, float* indices, float* width_offset, float* height_offset, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned long numThreads = blockDim.x * gridDim.x;
const unsigned long total_pixels = patch_width * patch_height * num_colors * num_images;
unsigned long ind, pos;
unsigned long image_id, dest_row, dest_col, color, source_row, source_col;
for (unsigned long i = idx; i < total_pixels; i += numThreads) {
ind = i;
image_id = ind % num_images; ind /= num_images;
dest_col = ind % patch_width; ind /= patch_width;
dest_row = ind % patch_height; ind /= patch_height;
color = ind % num_colors;

source_row = int(height_offset[image_id]) + dest_row;
source_col = int(width_offset[image_id]) + dest_col;
pos = img_width * img_height * num_colors * (int)indices[image_id] + img_width * img_height * color + img_width * source_row + source_col;
patches[i] = images[pos];
}
}