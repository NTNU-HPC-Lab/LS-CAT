#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftMaxCrossEntropyRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < height; i += numThreads) {
target[i] = -__logf(mat[height * (int)labels[i] + i] + tiny);
}
}