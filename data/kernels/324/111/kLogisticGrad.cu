#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogisticGrad(float* mat, float* targets, float* out_grad, unsigned int numEls) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < numEls; i += numThreads) {
out_grad[i] = (targets[i] < 0) ? 0 : (mat[i] - targets[i]);
}
}