#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
const unsigned int idx = threadIdx.x;
const unsigned int numThreads = blockDim.x;
const int off = blockIdx.x * numdims;

for (unsigned int target_ind = idx; target_ind < numdims; target_ind += numThreads) {
const int block = target_ind / blocksize;
target[off + target_ind] = source[off + block * blocksize];
}
}