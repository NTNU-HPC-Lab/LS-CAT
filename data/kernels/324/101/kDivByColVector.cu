#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = mat[i] / vec[i % height];
}
}