#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBoundingBoxLogisticGrad( float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
const int color = blockIdx.z;
/*
const int numXBlocksPerImage = DIVUP(width, blockDim.x);
const int image_id = blockIdx.x / numXBlocksPerImage;
const int col = (blockIdx.x % numXBlocksPerImage) * blockDim.x + threadIdx.x;
const int row = blockIdx.y * blockDim.y + threadIdx.y;
*/
const int image_id = threadIdx.x;
const int col = blockIdx.x;
const int row = blockIdx.y;

int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;
if (col < width && row < height && image_id < size && color < depth) {
int src_image_id = (int)indices[image_id];
int src_col = (int)(scale_width * col);
int src_row = (int)(scale_height * row);

int start = seg[src_image_id];
int end = seg[src_image_id + 1];
int x1, y1, x2, y2, l, inside;
for (int box_id = start; box_id < end; box_id++) {
l = label[box_id];
x1 = bbox[box_id << 2] - width_offset[image_id];
y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2) ? 1:0;
num_bboxes += inside;
num_bboxes_of_this_depth += (l == color) ? 1: 0;
num_bboxes_of_this_depth_inside += (inside == 1 && l == color) ? 1: 0;
}
}
unsigned long i = image_id + size * (col + width * (row + height * color));
__syncthreads();
if (col < width && row < height && image_id < size && color < depth) {
if (num_bboxes > 0) {
grad[i] = (num_bboxes_of_this_depth_inside > 0) ? (mat[i] - 1) : 0;
} else {
grad[i] = (num_bboxes_of_this_depth > 0) ? mat[i] : 0;
}
}
}