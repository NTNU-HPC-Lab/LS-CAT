#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExp(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = __expf(mat[i]);
}