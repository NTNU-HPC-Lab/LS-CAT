#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kWriteRowsMult(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float alpha, float beta) {
int c = blockIdx.y;
int src_module_id = blockIdx.x;
int dst_module_id = module_id_offset + blockIdx.x;

data += num_images * (src_module_id + c * num_modules_batch);
target += num_images * (dst_module_id + c * num_modules);

for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
target[im] = alpha * target[im] + beta * data[im];
}
}