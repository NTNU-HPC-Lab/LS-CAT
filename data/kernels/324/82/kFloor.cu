#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kFloor(float* mat, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) target[i] = floor(mat[i]);
}