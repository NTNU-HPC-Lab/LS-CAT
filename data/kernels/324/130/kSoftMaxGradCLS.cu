#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftMaxGradCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width*height; i += numThreads) {
target[i] = mat[i] - (labels[(int)indices[i % height]] == i / height ? 1 : 0);
}
}