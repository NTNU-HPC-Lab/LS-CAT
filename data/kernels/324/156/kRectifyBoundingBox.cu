#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRectifyBoundingBox( float* boxes, float* width_offset, float* height_offset, float* flip, int num_images, int patch_width, int patch_height, int num_locs) {

for (int loc_id = blockIdx.x; loc_id < num_locs; loc_id += gridDim.x) {
float *xmin_block = boxes + num_images * loc_id,
*ymin_block = boxes + num_images * (loc_id + num_locs),
*xmax_block = boxes + num_images * (loc_id + num_locs * 2),
*ymax_block = boxes + num_images * (loc_id + num_locs * 3);

for (int image_id = threadIdx.x; image_id < num_images; image_id += blockDim.x) {
float xmin = (flip[image_id] > 0.5) ? (256.0/patch_width - xmax_block[image_id]) : xmin_block[image_id],
xmax = (flip[image_id] > 0.5) ? (256.0/patch_width - xmin_block[image_id]) : xmax_block[image_id],
ymin = ymin_block[image_id],
ymax = ymax_block[image_id],
wo = width_offset[image_id],
ho = height_offset[image_id];

xmin_block[image_id] = xmin - wo / patch_width;
xmax_block[image_id] = xmax - wo / patch_width;

ymin_block[image_id] = ymin - ho / patch_height;
ymax_block[image_id] = ymax - ho / patch_height;
}
}
}