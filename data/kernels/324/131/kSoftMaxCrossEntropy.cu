#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width; i += numThreads) {
target[i] = -__logf(mat[height * i + (int)labels[i]] + tiny);
}
}