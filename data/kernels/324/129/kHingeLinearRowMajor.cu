#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kHingeLinearRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
int image_id = blockIdx.x * blockDim.x + threadIdx.x;
if (image_id < height) {
mat += image_id;
target += image_id;
const int correct_label = (int)labels[image_id];
const float correct_label_score = mat[correct_label * height];
float sum = 0;
for (unsigned int i = 0; i < width; i++) {
float diff = margin + mat[i*height] - correct_label_score;
float grad = (diff > 0) ? 1 : 0;
target[i*height] = (i == correct_label) ? 0 : grad;
sum             += (i == correct_label) ? 0 : grad;
}
target[correct_label * height] = -sum;
}
}