#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SimpleClone( const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox )
{
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt*yt+xt;
if (yt < ht and xt < wt and mask[curt] > 127.0f) {
const int yb = oy+yt, xb = ox+xt;
const int curb = wb*yb+xb;
if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
output[curb*3+0] = target[curt*3+0];
output[curb*3+1] = target[curt*3+1];
output[curb*3+2] = target[curt*3+2];
}
}
}