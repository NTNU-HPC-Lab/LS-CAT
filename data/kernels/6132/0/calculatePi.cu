#include "hip/hip_runtime.h"
#include "includes.h"
/**
* calculate pi
*/
// For the CUDA runtime routines (prefixed with "cuda_")
//Tiempo

#define NUMTHREADS 10240
#define ITERATIONS 1e12

/**
* CUDA Kernel Device code
*
*/
/*****************************************************************************/



/******************************************************************************
* Host main routine
*/
__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{   long int initialIteration, endIteration;
long int i = 0;
double piPartial;

//TamanioBloque*IdBloque + IdHilo
int index = (blockDim.x * blockIdx.x) + threadIdx.x;

initialIteration = (iterations/totalThreads) * index;
endIteration = initialIteration + (iterations/totalThreads) - 1;

i = initialIteration;
piPartial = 0;

do{
piPartial = piPartial + (double)(4.0 / ((i*2)+1));
i++;
piPartial = piPartial - (double)(4.0 / ((i*2)+1));
i++;
}while(i < endIteration);

piTotal[index] = piPartial;

__syncthreads();
if(index == 0){
for(i = 1; i < totalThreads; i++)
piTotal[0] = piTotal[0] + piTotal[i];
}
}