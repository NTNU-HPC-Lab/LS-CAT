#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateImg(unsigned char * data, unsigned char * img, unsigned char * tabDepth, int4 * _tabParents, int i, int tailleTab) {
int thx = blockIdx.x * blockDim.x + threadIdx.x;
int thy = blockIdx.y * blockDim.y + threadIdx.y;
int ThId = thy * tailleTab + thx;
int nbPar = 0;

if(data[ThId] == 0 && tabDepth[ThId] == i  && i != 1) {

if(_tabParents[ThId].x != -1) nbPar ++;
if(_tabParents[ThId].y != -1) nbPar ++;
if(_tabParents[ThId].z != -1) nbPar ++;
if(_tabParents[ThId].w != -1) nbPar ++;

data[ThId] = (data[_tabParents[ThId].x] + data[_tabParents[ThId].y] + data[_tabParents[ThId].z] + data[_tabParents[ThId].w]) / nbPar;

img[ThId] = data[ThId];
}

}