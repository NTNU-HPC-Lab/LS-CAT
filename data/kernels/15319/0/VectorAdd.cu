#include "hip/hip_runtime.h"
#include "includes.h"
// cuda ÇÁ·Î±×·¡¹Ö Ã¹ ½ÃÀÛÀ» ÇÏ±â À§ÇÑ ±âº» default ÄÚµå¸¦ °¡Á®¿Ô´Ù.
// ¼³¸íÀº ¾ÆÁ÷ÀÌ´Ï ¿ì¼± c++½ºÅ¸ÀÏ ÄÚµùÀÌ³ª ÀÍÈ÷°í, SIZE ¼ýÀÚ¸¦ ¹Ù²ã°¡¸ç ½ÇÇàÇÑ °á°ú¸¦ »ìÆìº¸ÀÚ
// µüÈ÷ ¿©±â¼­ ¹è¿ï °ÍÀº ¾ø°í cuda_main2.cu ºÎÅÍ ÇÏ³ª¾¿ Â÷±ÙÂ÷±Ù Â¤¾îº¼ ¿¹Á¤

#define SIZE 1024

// __global__À» ÅëÇØ¼­ Ä¿³ÎÀÓÀ» Ç¥½ÃÇÑ´Ù. host¿¡¼­ È£ÃâµÈ´Ù.

__global__ void VectorAdd(int *a, int *b, int *c, int n) {
// ¼ö¸¹Àº ½º·¹µå°¡ µ¿½Ã¿¡ Ã³¸®ÇÑ´Ù.
// µû¶ó¼­ threadIdx(½º·¹µå ÀÎµ¦½º)¸¦ ÅëÇØ¼­ ½º·¹µåµéÀ» ±¸º°ÇÑ´Ù.
int i = threadIdx.x;

printf("threadIdx.x : %d, n : %d\n", i, n);

for (i = 0; i < n; i++) {
c[i] = a[i] + b[i];
printf("%d = %d + %d\n", c[i], a[i], b[i]);
}
}