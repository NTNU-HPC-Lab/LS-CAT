#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void generate2DGaussian(double * output, double sigma, int sz, bool normalize) {

/*x and y coordinates of thread in kernel. The gaussian filters are
*small enough for the kernel to fit into a single thread block of sz*sz*/
const int colIdx = threadIdx.x;
const int rowIdx = threadIdx.y;
int linearIdx = rowIdx*sz + colIdx;

/*calculate distance from centre of filter*/
int distx = abs(colIdx - sz/2);
int disty = abs(rowIdx - sz/2);

output[linearIdx] = exp(-(pow((double)(distx), 2.0)+pow((double)(disty), 2.0))/(2*(pow(sigma, 2.0))));

if(normalize==true) {

/*wait until all threads have assigned a value to their index in the output array*/
__syncthreads();

int i, j;
double sum=0.0;

for(i=0; i<sz; i++) {
for(j=0; j<sz; j++) {
sum += output[i*sz + j];
}
}

/*Let all threads calculate the sum before changing the value of the output array*/
__syncthreads();

output[linearIdx]/=sum;
}
}
__global__ void getDoG(double * output, double sigma, double sigmaratio) {

int sz = ceil(sigma*3) * 2 + 1;
int linearIdx = threadIdx.y*sz + threadIdx.x;
if(linearIdx>=sz*sz) return;

__shared__ double g1[900];
__shared__ double g2[900];

generate2DGaussian(g1, sigma, sz, true);
generate2DGaussian(g2, sigma*sigmaratio, sz, true);

__syncthreads();

output[linearIdx] = g2[linearIdx]-g1[linearIdx];
}