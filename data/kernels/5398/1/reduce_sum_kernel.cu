#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_sum_kernel(const float *input, float *d_out,  int size) {
int tid         = threadIdx.x;                              // Local thread index
int myId        = blockIdx.x*(blockDim.x*2) + threadIdx.x;   // Global thread index

extern __shared__ float tempsum[]; //shared memory

// --- Loading data to shared memory. All the threads contribute to loading the data to shared memory.
tempsum[tid] = (myId < size) ? input[myId] + input[myId+blockDim.x] : 0.0f;

// --- make sure that all the shared memory loads have been completed
__syncthreads();

// --- Reduction in shared memory. Only half of the threads contribute to reduction.
for (unsigned int s=blockDim.x/2; s>0; s>>=1){
if (tid < s) { tempsum[tid] += tempsum[tid + s]; }
// --- make sure that all memory operations have been completed
__syncthreads();
}

if (tid == 0) {
d_out[blockIdx.x] = tempsum[0];
}
}