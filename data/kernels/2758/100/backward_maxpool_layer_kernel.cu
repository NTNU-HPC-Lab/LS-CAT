#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
int h = (in_h + pad - size) / stride_y + 1;
int w = (in_w + pad - size) / stride_x + 1;
int c = in_c;
int area_x = (size - 1) / stride_x;
int area_y = (size - 1) / stride_y;

int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= n) return;

int index = id;
int j = id % in_w;
id /= in_w;
int i = id % in_h;
id /= in_h;
int k = id % in_c;
id /= in_c;
int b = id;

int w_offset = -pad / 2;
int h_offset = -pad / 2;

float d = 0;
int l, m;
for(l = -area_y; l < area_y+1; ++l){
for(m = -area_x; m < area_x+1; ++m){
int out_w = (j-w_offset)/stride_x + m;
int out_h = (i-h_offset)/stride_y + l;
int out_index = out_w + w*(out_h + h*(k + c*b));
int valid = (out_w >= 0 && out_w < w &&
out_h >= 0 && out_h < h);
d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
}
}
prev_delta[index] += d;
}