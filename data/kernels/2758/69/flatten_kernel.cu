#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i >= N) return;
int in_s = i%spatial;
i = i/spatial;
int in_c = i%layers;
i = i/layers;
int b = i;

int i1 = b*layers*spatial + in_c*spatial + in_s;
int i2 = b*layers*spatial + in_s*layers +  in_c;

if (forward) out[i2] = x[i1];
else out[i1] = x[i2];
}