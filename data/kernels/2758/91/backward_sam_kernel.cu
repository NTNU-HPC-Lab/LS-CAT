#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_sam_kernel(float *in_w_h_c_delta, int size, int channel_size, float *in_scales_c, float *out_from_delta, float *in_from_output, float *out_state_delta)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
out_state_delta[index] += in_w_h_c_delta[index] * in_from_output[index]; // l.delta * from  (should be divided by channel_size?)
out_from_delta[index] += in_scales_c[index] * in_w_h_c_delta[index]; // input * l.delta

//out_state_delta[index] += in_w_h_c_delta[index];
//out_from_delta[index] = in_w_h_c_delta[index];
}
}