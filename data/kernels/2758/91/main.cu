#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "backward_sam_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *in_w_h_c_delta = NULL;
hipMalloc(&in_w_h_c_delta, XSIZE*YSIZE);
int size = XSIZE*YSIZE;
int channel_size = XSIZE*YSIZE;
float *in_scales_c = NULL;
hipMalloc(&in_scales_c, XSIZE*YSIZE);
float *out_from_delta = NULL;
hipMalloc(&out_from_delta, XSIZE*YSIZE);
float *in_from_output = NULL;
hipMalloc(&in_from_output, XSIZE*YSIZE);
float *out_state_delta = NULL;
hipMalloc(&out_state_delta, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
backward_sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c_delta,size,channel_size,in_scales_c,out_from_delta,in_from_output,out_state_delta);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
backward_sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c_delta,size,channel_size,in_scales_c,out_from_delta,in_from_output,out_state_delta);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
backward_sam_kernel<<<gridBlock,threadBlock>>>(in_w_h_c_delta,size,channel_size,in_scales_c,out_from_delta,in_from_output,out_state_delta);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}