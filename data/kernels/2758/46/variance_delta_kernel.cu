#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= filters) return;
int j,k;
variance_delta[i] = 0;
for(j = 0; j < batch; ++j){
for(k = 0; k < spatial; ++k){
int index = j*filters*spatial + i*spatial + k;
variance_delta[i] += delta[index]*(x[index] - mean[i]);
}
}
variance_delta[i] *= -.5 * powf(variance[i] + .000001f, (float)(-3./2.));
}