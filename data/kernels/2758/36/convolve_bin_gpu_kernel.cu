#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
return ~(a^b) & 0b1;
}
__device__ __host__ static inline unsigned char get_bit(unsigned char const*const src, size_t index) {
size_t src_i = index / 8;
int src_shift = index % 8;
unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
//unsigned char val = (src[src_i] & (1 << (8 - src_shift))) > 0;
return val;
}
__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;

int fil;
// filter index
//for (fil = 0; fil < n; ++fil)
int chan, y, x, f_y, f_x;
// channel index
//for (chan = 0; chan < in_c; ++chan)
// input - y
//for (y = 0; y < in_h; ++y)
// input - x
//for (x = 0; x < in_w; ++x)
x = index % in_w;
int index2 = index / in_w;
y = index2 % in_h;
fil = index2 / in_h;
//if (fil < n)    // (1-6 for one BLOCK)
{
//float mean_val = mean_arr_gpu[fil];
int const output_index = fil*in_w*in_h + y*in_w + x;
int sum = 0;
int good_val = 0;

int min_index = blockIdx.x*blockDim.x;
int min_fil = (min_index / in_w) / in_h;
int max_index = (blockIdx.x+1)*blockDim.x - 1;
int max_fil = (max_index / in_w) / in_h;

__shared__ uint32_t weights_shared[3*3*1024*6/32 + 1];  // 7 KB (6 filters) - use (new_lda) for size calculation
//const int weights_size = size*size*in_c/8;
const int weights_size = size*size*in_c / 32 + 1;

for (int tmp_fil = min_fil; tmp_fil <= max_fil; tmp_fil++) {
for (int s = threadIdx.x; s < weights_size; s += blockDim.x) {
//weights_shared[s + (tmp_fil - min_fil)*new_lda / 8] = ((uint8_t *)weights)[tmp_fil*new_lda / 8 + s];
weights_shared[s + (tmp_fil - min_fil)*new_lda/32] = ((uint32_t *)weights)[tmp_fil*new_lda / 32 + s];
}
}
__syncthreads();

for (chan = 0; chan < in_c; ++chan)
{
//int const weights_pre_index = fil*in_c*size*size + chan*size*size;
//int const weights_pre_index = fil*new_lda + chan*size*size;
int const input_pre_index = chan*in_w*in_h;

__shared__ uint32_t input_shared[416*416/32 + 1];   // 21.2 KB bytes (for input size 832x832)
const int input_shared_size = in_w*in_h / 32 + 1;
const int add_input_index = input_pre_index % 32;
__syncthreads();    // why??? but is required

for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
input_shared[s] = ((uint32_t *)input)[input_pre_index / 32 + s];
}
__syncthreads();

/*
__shared__ uint8_t input_shared[208 * 208 / 8 + 1];   // 5.4 KB bytes (for input size 416x416)
const int input_shared_size = in_w*in_h / 8 + 1;
const int add_input_index = input_pre_index % 8;
__syncthreads();

for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
((uint8_t *)input_shared)[s] = ((uint8_t *)input)[input_pre_index / 8 + s];
}
__syncthreads();
*/
//int src_index = -1;
//uint32_t input_byte;

if (fil < n)    // (1-6 for one BLOCK)
{
// filter - y
for (f_y = 0; f_y < size; ++f_y)
{
int input_y = y + f_y - pad;
// filter - x
for (f_x = 0; f_x < size; ++f_x)
{
int input_x = x + f_x - pad;
if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

//int input_index = input_pre_index + input_y*in_w + input_x;
//int weights_index = weights_pre_index + f_y*size + f_x;
//int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
//int weights_index = fil*new_lda + chan*size*size + f_y*size + f_x;

//uint8_t in_bit = get_bit((uint8_t *)input, input_index);
//uint8_t w_bit = get_bit((uint8_t *)weights, weights_index);

//int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
int weights_shared_index = (fil - min_fil)*new_lda + chan*size*size + f_y*size + f_x;
//uint8_t in_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);
uint8_t w_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);

//int input_index = input_pre_index + input_y*in_w + input_x;
int input_shared_index = /*input_pre_index +*/ input_y*in_w + input_x + add_input_index;
uint8_t in_bit = get_bit((uint8_t *)input_shared, input_shared_index);
/*
int new_src_index = input_shared_index / 32;
int src_shift = input_shared_index % 32;
//if (new_src_index != src_index)
{
src_index = new_src_index;
input_byte = ((uint32_t *)input_shared)[src_index];
}
uint8_t in_bit = (input_byte & (1 << src_shift)) >> src_shift;
*/

int res = xnor_bit1(in_bit, w_bit);
sum += res;
good_val++;

//sum += input[input_index] *weights[weights_index];

}
}
}
// l.output[filters][width][height] +=
//        state.input[channels][width][height] *
//        l.weights[filters][channels][filter_width][filter_height];
//output[output_index] += sum;
}
sum = sum - (good_val - sum);
//output[output_index] = sum * mean_arr_gpu[fil]; // atoimcAdd for inter-BLOCK sum
atomicAdd(&output[output_index], sum * mean_arr_gpu[fil]);
}

}