#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
__shared__ uint32_t tmp[33 * 32];   // misaligned_array[32x32]
const int w_align = 33;
//const int shared_size = w_align * 32;

//l.bit_align - algined (n) by 32
//new_ldb - aligned (k) by 256

const int src_w_align = src_w + (32 - src_w % 32);
//const int src_h_align = src_h + (32 - src_h % 32);

const int warps_in_width = src_w_align / 32;
//const int warps_in_height = src_h_align / 32;



const int local_x = threadIdx.x % 32;   // index % 32;
const int local_x_index = threadIdx.x / 32; // index / 32;
const int local_y = local_x_index % 32;

//#pragma unroll TRANS_LOOP
//for (int i = 0; i < TRANS_LOOP; ++i)
{
const int global_index = blockIdx.x;// blockIdx.x*TRANS_LOOP + i;// local_x_index / 32;
const int global_x_index = global_index % warps_in_width;
const int global_y_index = global_index / warps_in_width;

const int global_x = global_x_index * 32 + local_x;
const int global_y = global_y_index * 32 + local_y;

uint32_t val = 0;
if (global_x < src_w && global_y < src_h) {
val = src[global_y * src_align + global_x];
}
//dst[global_x * dst_align / 32 + global_y] = val;
//tmp[local_y * 32 + local_x] = val;

tmp[local_x * w_align + local_y] = val;
__syncthreads();
val = tmp[local_y * w_align + local_x];

const int new_global_x = global_y_index * 32 + local_x;
const int new_global_y = global_x_index * 32 + local_y;

if (new_global_x < src_h && new_global_y < src_w) {
dst[new_global_y * (dst_align / 32) + new_global_x] = val;
}
}
}