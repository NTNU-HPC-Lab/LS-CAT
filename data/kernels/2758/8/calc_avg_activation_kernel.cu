#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int xy = i % size;
int b = i / size;

if (i < size*batches) {
dst[i] = 0;
for (int c = 0; c < channels; ++c) {
dst[i] += src[xy + size*(c + channels*b)];
}
dst[i] = dst[i] / channels;
}
}