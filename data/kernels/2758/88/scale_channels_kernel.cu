#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
out[index] = in_w_h_c[index] * scales_c[index / channel_size];
}
}