#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
float val = input[index];
if (isnan(val) || isinf(val))
input[index] = 1.0f / index;  // pseudo random value
}
}