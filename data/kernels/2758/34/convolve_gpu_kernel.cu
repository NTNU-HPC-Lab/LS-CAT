#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;

int fil;
// filter index
//for (fil = 0; fil < n; ++fil)
int chan, y, x, f_y, f_x;
// channel index
//for (chan = 0; chan < in_c; ++chan)
// input - y
//for (y = 0; y < in_h; ++y)
// input - x
//for (x = 0; x < in_w; ++x)
x = index % in_w;
int index2 = index / in_w;
y = index2 % in_h;
fil = index2 / in_h;
if (fil < n)
{

int const output_index = fil*in_w*in_h + y*in_w + x;
float sum = 0;

for (chan = 0; chan < in_c; ++chan)
{
int const weights_pre_index = fil*in_c*size*size + chan*size*size;
int const input_pre_index = chan*in_w*in_h;

// filter - y
for (f_y = 0; f_y < size; ++f_y)
{
int input_y = y + f_y - pad;
// filter - x
for (f_x = 0; f_x < size; ++f_x)
{
int input_x = x + f_x - pad;
if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

int input_index = input_pre_index + input_y*in_w + input_x;
int weights_index = weights_pre_index + f_y*size + f_x;

sum += input[input_index] * weights[weights_index];

}
}
// l.output[filters][width][height] +=
//        state.input[channels][width][height] *
//        l.weights[filters][channels][filter_width][filter_height];
//output[output_index] += sum;
}
output[output_index] = sum;
}

}