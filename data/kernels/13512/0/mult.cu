#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright (C) 2009-2012 Fraunhofer SCAI, Schloss Birlinghoven, 53754 Sankt Augustin, Germany;
all rights reserved unless otherwise stated.

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston,
MA 02111-1307 USA
*/




/** Index function to address the two-dimensional arrays
Q and R

Matrices are stored in column-major order (like Fortran).

i is the row, j is the column (index starts at 1)
ld is the number of elements for each column
*/

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

/* ---------------------------------------------------------------------- */

/*  Tuning can be done by different block sizes. */

#define BLOCK1 64

// 8800 GT:    128 x 1
// C1060:      128 x 1
#define BLOCK1X    64
#define BLOCK1Y    8

// 8800 GT:      64 x 4
// C1060:        64 x 8
#define BLOCK2X   512
#define BLOCK2Y   1
/* ---------------------------------------------------------------------- */

/** Kernel for matrix-vector multiplication

R(k,k:n) = matmulv( Q(1:m,k:n), Q(1:m) )

Same as this BLAS-2 call:

call sgemv('T', m, n-k+1, 1.0, Q(1,k), M, Q(1,k), 1, 0.0, R(k,k), N)

The threads in x-dimension are used for parallelization of
the dot products, the threads in y-dimension compute different
elements of the result vector.

Each thread (t1,t2)  will be responsible for BLOCK1X columns and BLOCK1Y
rows of the matrix Q.
*/


/* ---------------------------------------------------------------------- */

/** This kernel scales the row k of the matrix R

R(k,k:n) = R(k,k:n) * S
*/


/* ---------------------------------------------------------------------- */

/** This kernel scales the column k of the matrix Q.

Q(1:m,k) = Q(1:m,k) * S
*/


/* ---------------------------------------------------------------------- */

/** This kernel updates the matrix Q by a product of two vectors.

Q(1:m,k+1:n) -= R(k,k+1:n) * Q(1:m,k)

same as this BLAS-2 call:

call sger(M, N-K, -1.0, Q(1,K), 1, R(K,K+1), N, Q(1,K+1), M)

Each thread (t1,t2)  will be responsible for BLOCK2X columns and BLOCK2Y
rows of the matrix Q.
*/


/* ---------------------------------------------------------------------- */

/**  QR factorization of a matrix

@param[in]      m is number of rows for Q and R
@param[in]      n is number of columns for Q and R
@param[in,out]  Q is a matrix of size m x n, column major order
@param[out]     R is a matrix of size m x n, column major order

@returns 0 if successful

Q(in) = Q(out) * R, where Q(out) is orthonormal and R upper-triangular
*/

__global__ void mult(float* Q, float* R, int m, int n, int k)
{
__shared__ float RS[BLOCK1Y][BLOCK1X];
__shared__ float QK[BLOCK1Y];

int tid1 = threadIdx.x;
int tid2 = threadIdx.y;

int i = blockIdx.x * BLOCK1Y + tid2 + k;

float S = 0.0f;

if (i < k or i > n) return;

for (int j = tid1+1; j <= m; j+=BLOCK1X) {
if (tid1 == 0) QK[tid2] = Q[IDX2F(j,k,m)];
__syncthreads();
S += QK[tid2] * Q[IDX2F(j,i,m)];
}

// thread writes result in shared array RS

RS[tid2][tid1] = S;

int NT = BLOCK1X;

while (NT > 1) {
// first half of threads sums up
__syncthreads();
NT = NT >> 1 ;
if (tid1 < NT) {
RS[tid2][tid1] += RS[tid2][tid1+NT];
}
}

// now thread 0 writes the result

if (tid1 == 0) {
R[IDX2F(k,i,n)] = RS[tid2][0];
}
}