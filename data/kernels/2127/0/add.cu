#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y) {

// Calculate the starting value for the for loop's index
int index = ( blockDim.x * blockIdx.x ) + threadIdx.x;

// Calculate the stride between elements of the arrays
int stride = blockDim.x * gridDim.x;

// Add the elements from array x and array y within the block
for (int i = index; i < n; i += stride)
// Store the result in y
y[i] = x[i] + y[i];
}