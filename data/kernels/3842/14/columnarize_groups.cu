#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
const auto thread_index =
threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
if (thread_index >= row_count) {
return;
}
auto read_ptr = rowwise_buffer + thread_index * row_size;
auto col_base = columnar_buffer;
for (size_t i = 0; i < col_count; ++i) {
switch (col_widths[i]) {
case 8: {
int64_t* write_ptr = reinterpret_cast<int64_t*>(col_base) + thread_index;
*write_ptr = *reinterpret_cast<const int64_t*>(read_ptr);
} break;
case 4: {
int32_t* write_ptr = reinterpret_cast<int32_t*>(col_base) + thread_index;
*write_ptr = *reinterpret_cast<const int32_t*>(read_ptr);
} break;
default:;
}
col_base += col_widths[i] * row_count;
read_ptr += col_widths[i];  // WARN(miyu): No padding!!
}
}