#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
const int32_t step = blockDim.x * gridDim.x;
for (int32_t i = start; i < entry_count; i += step) {
if (VALID_POS_FLAG == pos_buff[i]) {
pos_buff[i] = !i ? 0 : count_buff[i - 1];
}
}
}