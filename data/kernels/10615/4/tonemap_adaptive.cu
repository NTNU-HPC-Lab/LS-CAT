#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float adaptive_mapping(float k, float q, float val_pixel){
return 	(k*log(1 + val_pixel))/((100*log10(1 + maxLum)) * ( powf((log(2+8*(val_pixel/maxLum))), (log(q)/log(0.5)) ) )	);
}
__global__ void tonemap_adaptive(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float q, float k){
//printf("maxLum : %f\n", maxLum);
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

if(Row < height && Col < width) {
imageOut[(Row*width+Col)*3+BLUE] = adaptive_mapping(k, q, imageIn[(Row*width+Col)*3+BLUE]);
imageOut[(Row*width+Col)*3+GREEN] = adaptive_mapping(k, q, imageIn[(Row*width+Col)*3+GREEN]);
imageOut[(Row*width+Col)*3+RED] = adaptive_mapping(k, q, imageIn[(Row*width+Col)*3+RED]);
}
}