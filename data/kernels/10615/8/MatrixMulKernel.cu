#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P,int width){
__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

float Pvalue = 0;
//printf("%f\n", width/TILE_WIDTH );
for (int i = 0; i < width/TILE_WIDTH; ++i){
//printf("%d\n", i );

Mds[ty][tx] = d_M[row*width + i*TILE_WIDTH + tx];
Nds[ty][tx] =  d_N[(i*TILE_WIDTH + ty)*width + col];
__syncthreads();

for (int j = 0; j < TILE_WIDTH; ++j){
Pvalue += Mds[ty][j] * Nds[j][tx];
}
__syncthreads();
}
d_P[row*width + col] = Pvalue;
}