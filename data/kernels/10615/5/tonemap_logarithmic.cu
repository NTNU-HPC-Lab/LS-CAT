#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float logarithmic_mapping(float k, float q, float val_pixel){
return (log10(1 + q * val_pixel))/(log10(1 + k * maxLum));
}
__global__ void tonemap_logarithmic(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float q, float k){
//printf("maxLum : %f\n", maxLum);
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

if(Row < height && Col < width) {
imageOut[(Row*width+Col)*3+BLUE] = logarithmic_mapping(k, q, imageIn[(Row*width+Col)*3+BLUE]);
imageOut[(Row*width+Col)*3+GREEN] = logarithmic_mapping(k, q, imageIn[(Row*width+Col)*3+GREEN]);
imageOut[(Row*width+Col)*3+RED] = logarithmic_mapping(k, q, imageIn[(Row*width+Col)*3+RED]);
}
}