#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P,int width){
int Row = blockIdx.y*blockDim.y + threadIdx.y;
int Col = blockIdx.x*blockDim.x + threadIdx.x;

if ((Row < width)&&(Col < width)){
float Pvalue = 0;
for (int i = 0; i < width; ++i){
Pvalue += d_M[Row*width+i]*d_N[i*width+Col];
}
d_P[Row*width + Col] = Pvalue;
}
}