#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if((row < height) && (col < width)){
imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587
+ imageInput[(row*width+col)*3+BLUE]*0.114;
}
}