#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float gamma_correction(float f_stop, float gamma, float val)
{
return powf((val*powf(2,f_stop)),(1.0/gamma));
}
__global__ void tonemap_gamma(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float f_stop, float gamma)
{
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

if(Row < height && Col < width) {
imageOut[(Row*width+Col)*3+BLUE] = gamma_correction(f_stop, gamma, imageIn[(Row*width+Col)*3+BLUE]);
imageOut[(Row*width+Col)*3+GREEN] = gamma_correction(f_stop, gamma, imageIn[(Row*width+Col)*3+GREEN]);
imageOut[(Row*width+Col)*3+RED] = gamma_correction(f_stop, gamma, imageIn[(Row*width+Col)*3+RED]);
}
}