#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_maximum_kernel(float *array, int *mutex, unsigned int n, int blockSize){
unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
unsigned int stride = gridDim.x*blockDim.x;
unsigned int offset = 0;

extern	__shared__ float cache[];

float temp = -1.0;
while(index + offset < n){
temp = fmaxf(temp, array[index + offset]);

offset += stride;
}

cache[threadIdx.x] = temp;

__syncthreads();
// reduction
unsigned int i = blockDim.x/2;
while(i != 0){
if(threadIdx.x < i){
cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
}

__syncthreads();
i /= 2;
}

if(threadIdx.x == 0){
while(atomicCAS(mutex,0,1) != 0);  //lock
maxLum = fmaxf(maxLum, cache[0]);
atomicExch(mutex, 0);  //unlock
}
}