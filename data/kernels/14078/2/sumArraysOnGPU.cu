#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
//int i = threadIdx.x;
int i = blockIdx.x * blockDim.x + threadIdx.x; //general case
if (i < N) C[i] = B[i] + A[i];
}