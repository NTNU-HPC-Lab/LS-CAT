#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel(int n, float *arr){

volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;
if(idx >= n) return;

arr[idx] *= 2.0f;
}