#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_calc_gL( int layer_id, int *l, int *s, float *z_arr, float *a_arr, float *t_arr, float *gjl ){

volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];

if(idx >= neuron_count-1) return;

float z = z_arr[s[layer_id] + idx];
float tmp = 1 + expf(-z);
float f_deriv=expf(-z) / (tmp*tmp);

gjl[s[layer_id] + idx] = f_deriv*(a_arr[s[layer_id] + idx] - t_arr[idx]);
}