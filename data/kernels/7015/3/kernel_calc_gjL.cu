#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_calc_gjL( int layer_id, int *l, int *s, int *sw, float *z_arr, float *a_arr, float *t_arr, float *gjl, float *w_arr ){

volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];
int neuron_count_next = l[layer_id+1];

if(idx >= neuron_count-1) return;

//float f_deriv=expf(-z_arr[s[layer_id] + idx]) / powf((1 + expf(-z_arr[s[layer_id] + idx])),2.0f);
float z = z_arr[s[layer_id] + idx];
float tmp = 1 + expf(-z);
float f_deriv=expf(-z) / (tmp*tmp);


float sum = 0;
for (int k = 0; k < neuron_count_next-1; k++) {
sum += w_arr[sw[layer_id] + idx*(l[layer_id + 1] - 1) + k] * gjl[s[layer_id + 1] + k];
}

gjl[s[layer_id] + idx] = f_deriv*sum;
// printf("Kernelis %d - %.20f\n", s[layer_id] + idx, gjl[s[layer_id] + idx]);
}