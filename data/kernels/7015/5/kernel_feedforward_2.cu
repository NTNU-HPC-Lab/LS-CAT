#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_feedforward_2( int layer_id, int *l, int *s_ext, int *sw_ext, float *z_ext_arr, float *a_ext_arr, float *w_ext_arr ){

extern __shared__ int sm[];
float *sm_z = (float*)&sm[0];

int h = blockDim.x;
int h2 = blockDim.y;


int lidx = threadIdx.x;
int pidx = threadIdx.y;
int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];
int neuron_count_prev = l[layer_id-1];

//printf("layer = %d idx = %d count = %d\n", layer_id, idx, neuron_count-1);
if(idx >= neuron_count-1) return;

float z = 0;
int index0=sw_ext[layer_id-1];
int index1=s_ext[layer_id-1];
for(int k = pidx; k < neuron_count_prev; k+=h2){
z += w_ext_arr[index0 + k*(neuron_count - 1) + idx]*a_ext_arr[index1 + k];
}

sm_z[pidx*h + lidx] = z;


__syncthreads();

if(pidx == 0){
z = 0;
for(int i = 0; i < h2; i++)
z += sm_z[i*h + lidx];

z_ext_arr[s_ext[layer_id] + idx] = z;
float a = 1.0 / (1.0 + expf(-z));
a_ext_arr[s_ext[layer_id] + idx] = a;
}


// printf("index = %d z = %.5f\n", s[layer_id] + idx, z);
// printf("a = %.20f\n", a);
}