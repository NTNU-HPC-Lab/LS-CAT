#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_calc_gL_2( int layer_id, int *l, int *s_ext, float *z_ext_arr, float *a_ext_arr, float *t_arr, float *gjl_ext ){

volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];

if(idx >= neuron_count-1) return;

float z = z_ext_arr[s_ext[layer_id] + idx];
float tmp = 1 + expf(-z);
float f_deriv=expf(-z) / (tmp*tmp);

gjl_ext[s_ext[layer_id] + idx] = f_deriv*(a_ext_arr[s_ext[layer_id] + idx] - t_arr[idx]);
}