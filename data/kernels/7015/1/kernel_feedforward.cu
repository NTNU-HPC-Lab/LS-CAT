#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_feedforward( int layer_id, int *l, int *s, int *sw, float *z_arr, float *a_arr, float *w_arr ){
volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];
int neuron_count_prev = l[layer_id-1];

//printf("layer = %d idx = %d count = %d\n", layer_id, idx, neuron_count-1);
if(idx >= neuron_count-1) return;

float z = 0;
for(int k = 0; k < neuron_count_prev; k++){
z += w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k];
// printf("w_arr[%d] * a_arr[%d] = %.20f\n",
// 		sw[layer_id-1] + k*(neuron_count - 1) + idx ,
// 		s[layer_id-1] + k,
// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]);
// printf("%.10f * %.10f = %.10f\n", w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx ],
// 		a_arr[s[layer_id-1] + k],
// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]
// 	);

}

z_arr[s[layer_id] + idx] = z;
float a = 1.0 / (1.0 + expf(-z));
a_arr[s[layer_id] + idx] = a;
// printf("index = %d z = %.5f\n", s[layer_id] + idx, z);
// printf("a = %.20f\n", a);
}