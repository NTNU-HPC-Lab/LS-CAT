#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_weight_update( int layer_id, int *l, int *s, int *sw, float *z_arr, float *a_arr, float *t_arr, float *gjl, float *w_arr, float *dw_arr, float eta, float alpha ){

volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

int neuron_count = l[layer_id];
int neuron_count_next = l[layer_id+1];

if(idx >= neuron_count) return;

float a = a_arr[s[layer_id] + idx];
for(int k = 0; k < neuron_count_next-1; k++){

float grad=/*a_arr[s[layer_id] + idx]*/a*gjl[s[layer_id + 1] + k];

dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k]=
-eta*grad+
alpha*dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k];

w_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k]+=
dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k];
}
}