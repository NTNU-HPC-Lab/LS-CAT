#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_calc_gjL_2( int layer_id, int *l, int *s_ext, int *sw_ext, float *z_ext_arr, float *a_ext_arr, float *t_arr, float *gjl_ext, float *w_ext_arr ){

int idx = threadIdx.y + blockDim.y*blockIdx.y;
int h = blockDim.x;
int pidx = threadIdx.y;
int lidx = threadIdx.x;

extern __shared__ int sm[];
float *sm_g = (float*)&sm[0];


int neuron_count = l[layer_id];
int neuron_count_next = l[layer_id+1];

if(idx >= neuron_count-1) return;

float sum = 0;
for (int k = lidx; k < neuron_count_next-1; k+=h) {
sum += w_ext_arr[sw_ext[layer_id] + idx*(l[layer_id + 1] - 1) + k] * gjl_ext[s_ext[layer_id + 1] + k];
}

sm_g[pidx*h + lidx] = sum;

__syncthreads();

if(lidx == 0){
float z = z_ext_arr[s_ext[layer_id] + idx];
float tmp = 1 + expf(-z);
float f_deriv = expf(-z) / (tmp*tmp);

sum = 0;
for(int i = 0; i < h; i++)
sum += sm_g[pidx*h + i];


gjl_ext[s_ext[layer_id] + idx] = f_deriv*sum;
}
}