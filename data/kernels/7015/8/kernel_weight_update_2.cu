#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_weight_update_2( int layer_id, int *l, int *s_ext, int *sw_ext, float *z_ext_arr, float *a_ext_arr, float *t_arr, float *gjl_ext, float *w_ext_arr, float *dw_ext_arr, float eta, float alpha ){

int idx = threadIdx.y + blockDim.y*blockIdx.y;
int h = blockDim.x;
int pidx=threadIdx.x;


int neuron_count = l[layer_id];
int neuron_count_next = l[layer_id+1];

if(idx >= neuron_count) return;

float a = a_ext_arr[s_ext[layer_id] + idx];

int index0 = s_ext[layer_id + 1] + pidx;
int index1 = sw_ext[layer_id] + idx*(neuron_count_next - 1) + pidx;
for(int k = pidx; k < neuron_count_next-1; k+=h){

float grad = a*gjl_ext[index0];
index0 += h;
float dw = dw_ext_arr[index1] = -eta*grad + alpha*dw_ext_arr[index1];

w_ext_arr[index1] += dw;


index1 += h;

}
}