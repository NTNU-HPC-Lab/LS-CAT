#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernUpdatePi( const size_t numPoints, const size_t numComponents, double* logpi, double* Gamma ) {
int b = blockIdx.y * gridDim.x + blockIdx.x;
int comp = b * blockDim.x + threadIdx.x;
if(comp > numComponents) {
return;
}

__shared__ double A[1024];
A[comp] = logpi[comp] + log(Gamma[comp * numPoints]);
__syncthreads();

double sum = 0;
for(size_t k = 0; k < numComponents; ++k) {
sum += exp(A[k]);
}

logpi[comp] = A[comp] - log(sum);
}