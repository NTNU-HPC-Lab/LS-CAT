#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void devVecAdd(size_t pointDim, double* dest, double* src) {
for(size_t i = 0; i < pointDim; ++i) {
dest[i] += src[i];
}
}
__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, double* dest, double* src) {
// Called to standardize arrays to be a power of two

// Assumes a 2D grid of 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

if(i < numPoints) {
devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
}
}