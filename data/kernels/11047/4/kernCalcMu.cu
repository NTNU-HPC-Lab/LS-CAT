#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernCalcMu( const size_t numPoints, const size_t pointDim, const double* X, const double* loggamma, const double* GammaK, double* dest ) {
// Assumes a 2D grid of 1024x1 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;
if(i >= numPoints) {
return;
}

const double a = exp(loggamma[i]) / exp(*GammaK);
const double* x = & X[i * pointDim];
double* y = & dest[i * pointDim];

for(size_t i = 0; i < pointDim; ++i) {
y[i] = a * x[i];
}
}