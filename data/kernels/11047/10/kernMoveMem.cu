#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernMoveMem(const size_t numPoints, const size_t pointDim, const size_t s, double* A) {
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

// Before
// [abc......] [def......] [ghi......] [jkl......]

// shared memory
// [adgj.....]

// After
// [a..d..g..] [j........] [ghi......] [.........]

__shared__ double mem[1024];
mem[threadIdx.x] = A[s * i * pointDim];
__syncthreads();
A[i * pointDim] = mem[threadIdx.x];
}