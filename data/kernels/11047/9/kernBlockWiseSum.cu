#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, double* dest) {
// Assumes a 2D grid of 1024x1 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

// call repeatedly for each dimension where dest is assumed to begin at dimension d

__shared__ double blockSum[1024];

if(threadIdx.x >= numPoints) {
blockSum[threadIdx.x] = 0;
} else {
blockSum[threadIdx.x] = dest[i * pointDim];
}

__syncthreads();

// Do all the calculations in block shared memory instead of global memory.
for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
blockSum[threadIdx.x] += blockSum[threadIdx.x + s];
__syncthreads();
}

if(threadIdx.x == 0) {
// Just do one global write
dest[i * pointDim] = blockSum[0];
}
}