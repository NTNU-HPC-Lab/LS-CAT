#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernBlockWiseMax(const size_t numPoints, double* dest) {
// Assumes a 2D grid of 1024x1 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

__shared__ double blockMax[1024];

if(threadIdx.x >= numPoints) {
blockMax[threadIdx.x] = -INFINITY;
} else {
blockMax[threadIdx.x] = dest[i];
}

__syncthreads();

// Do all the calculations in block shared memory instead of global memory.
for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
if(blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
}
__syncthreads();
}

if(threadIdx.x == 0) {
// Just do one global write
dest[i] = blockMax[0];
}
}