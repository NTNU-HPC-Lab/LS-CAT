#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernExp(double* A, double* bias) {
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;
A[i] = exp(A[i] - *bias);
}