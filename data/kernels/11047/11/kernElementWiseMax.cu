#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernElementWiseMax(const size_t numPoints, double* dest, double* src) {
// Called to standardize arrays to be a power of two

// Assumes a 2D grid of 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

if(i < numPoints) {
if(dest[i] < src[i]) {
dest[i] = src[i];
}
}
}