#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_topp_id_val(int* topp_id_val_buf, int* topp_offset_buf, const int batch_size, const int vocab_size)
{
int tid = threadIdx.x;
int bid = blockIdx.x;

if(bid == 0)
{
for(int i = tid; i < batch_size + 1; i+= blockDim.x)
{
topp_offset_buf[i] = i * vocab_size;
}
}

while(tid < vocab_size)
{
topp_id_val_buf[bid * vocab_size + tid] = tid;
tid += blockDim.x;
}
}