#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GatherKernel(const float* params, int64_t num_features, int embed_size, int batch_size, int query_nnz, const int64_t* indices, float* ret) {
int tid = threadIdx.x, bid = blockIdx.x;

extern __shared__ int shmem_indices[];

// each CTA load one row of indices in the mini batch into shared memory
for (int i = tid; i < query_nnz; i += blockDim.x) {
shmem_indices[i] = indices[query_nnz * bid + i];
}
__syncthreads();

#pragma unroll
for (int i = 0; i < query_nnz; ++i) {
// printf("%d, %d, %d\n", bid, i, shmem_indices[i]);
ret[(bid * query_nnz + i) * embed_size + tid] =
params[(int64_t)shmem_indices[i] * embed_size + tid];
}
}