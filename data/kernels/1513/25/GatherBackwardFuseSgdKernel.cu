#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GatherBackwardFuseSgdKernel(const float* grads, int64_t num_features, int embed_size, int batch_size, int query_nnz, const int64_t* indices, float lr, float* params) {
int tid = threadIdx.x, bid = blockIdx.x;

extern __shared__ int shmem_indices[];

for (int i = tid; i < query_nnz; i += blockDim.x) {
shmem_indices[i] = indices[query_nnz * bid + i];
}
__syncthreads();

#pragma unroll
for (int i = 0; i < query_nnz; ++i) {
atomicAdd(&params[(int64_t)shmem_indices[i] * embed_size + tid],
-lr * grads[(bid * query_nnz + i) * embed_size + tid]);
}
}