#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OneHotKernel(const float* params, int64_t num_features, int embed_size, int batch_size, const int64_t* indices, float* ret) {
int tid = threadIdx.x, bid = blockIdx.x;

ret[bid * embed_size + tid] = params[(int64_t)indices[bid] * embed_size + tid];
}