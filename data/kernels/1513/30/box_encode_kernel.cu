#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void box_encode_kernel(float *targets_dx, float *targets_dy, float *targets_dw, float *targets_dh, float4 *boxes, float4 *anchors, float wx, float wy, float ww, float wh, size_t gt, size_t idxJump) {

int idx = blockIdx.x * blockDim.x + threadIdx.x;
size_t row_offset;
float anchors_x1, anchors_x2, anchors_y1, anchors_y2,
boxes_x1, boxes_x2, boxes_y1, boxes_y2, ex_w, ex_h,
ex_ctr_x, ex_ctr_y, gt_w, gt_h, gt_ctr_x, gt_ctr_y;

for (int i = idx; i < gt; i += idxJump){
row_offset = i;
anchors_x1 = anchors[row_offset].x;
anchors_y1 = anchors[row_offset].y;
anchors_x2 = anchors[row_offset].z;
anchors_y2 = anchors[row_offset].w;

boxes_x1 = boxes[row_offset].x;
boxes_y1 = boxes[row_offset].y;
boxes_x2 = boxes[row_offset].z;
boxes_y2 = boxes[row_offset].w;

ex_w = anchors_x2 - anchors_x1 + 1;
ex_h = anchors_y2 - anchors_y1 + 1;
ex_ctr_x = anchors_x1 + 0.5 * ex_w;
ex_ctr_y = anchors_y1 + 0.5 * ex_h;

gt_w = boxes_x2 - boxes_x1 + 1;
gt_h = boxes_y2 -  boxes_y1 + 1;
gt_ctr_x = boxes_x1 + 0.5 * gt_w;
gt_ctr_y = boxes_y1 + 0.5 * gt_h;

targets_dx[i] = wx * (gt_ctr_x - ex_ctr_x) / ex_w;
targets_dy[i] = wy * (gt_ctr_y - ex_ctr_y) / ex_h;
targets_dw[i] = ww * log(gt_w / ex_w);
targets_dh[i] = wh * log(gt_h / ex_h);
}

}