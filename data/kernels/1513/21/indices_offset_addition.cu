#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void indices_offset_addition(int64_t *indices, int64_t *offsets, int64_t *output_indices, int batch_size) {
const int fea_count = 26;
__shared__ int64_t smem_offsets[fea_count];

if (threadIdx.x < fea_count) {
smem_offsets[threadIdx.x] = offsets[threadIdx.x];
}
__syncthreads();

int start_idx = threadIdx.x + blockIdx.x * blockDim.x;
for (int i = start_idx; i < (batch_size * fea_count); i+=(gridDim.x * blockDim.x)) {
output_indices[i] = indices[i] + smem_offsets[i % fea_count];
}
}