#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void build_sequence_length_padding_offset(const int* sequence_length, const int batch_size, const int max_seq_len, int* valid_word_num, int* tmp_mask_offset)
{
// do cumulated sum
int total_seq_len = 0;
int cum_offset = 0;
int index = 0;
for(int i = 0; i < batch_size; i++)
{
const int seq_len = sequence_length[i];
for(int j = 0; j < seq_len; j++)
{
tmp_mask_offset[index] = cum_offset;
index++;
}
cum_offset += max_seq_len - seq_len;
total_seq_len += seq_len;
}
valid_word_num[0] = total_seq_len;
}