#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void topp_initialization_kernel(bool* finished, int* sequence_length, int* word_ids, int* topp_id_val_buf, int* topp_offset_buf, const int batch_size, const int vocab_size, const int start_id)
{
int tid = threadIdx.x;
int bid = blockIdx.x;

if(bid == 0)
{
for(int i = tid; i < batch_size + 1; i+= blockDim.x)
{
topp_offset_buf[i] = i * vocab_size;
}

for(int i = tid; i < batch_size; i+= blockDim.x)
{
finished[i] = false;
sequence_length[i] = 0;
word_ids[i] = start_id;
}
}

int index = tid + bid * blockDim.x;
while(index < batch_size * vocab_size)
{
topp_id_val_buf[index] = index % vocab_size;
index += blockDim.x * gridDim.x;
}
}