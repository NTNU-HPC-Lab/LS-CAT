#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void picaod_kernel(unsigned int *dev_v, long size, unsigned int *temp)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int offset = x + y * blockDim.x * gridDim.x;

atomicAdd(&(temp[0]), dev_v[offset]);

}