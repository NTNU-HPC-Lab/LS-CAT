#include "hip/hip_runtime.h"
#include "includes.h"


typedef float dtype;

#define N_ (8 * 1024 * 1024)
#define MAX_THREADS 256	   // threads per block
#define MAX_BLOCKS 64

#define MIN(x,y) ((x < y) ? x : y)


/* return the next power of 2 number that is larger than x */
__global__ void kernel5(dtype *g_idata, dtype *g_odata, unsigned int n)
{
__shared__  volatile dtype scratch[MAX_THREADS];
unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
unsigned int blockDimNew = blockDim.x * 2; 			// since the new blockDim will be halved for the loop iterations
unsigned int i = (bid * blockDimNew) + threadIdx.x;
unsigned int gridSize = blockDim.x * 2 * gridDim.x;

// each thread sums up 512 elements before storing into shared array
scratch[threadIdx.x] = 0;
while(i < n) {
scratch[threadIdx.x] += g_idata[i] + g_idata[i + blockDim.x];
i += gridSize;	//stride length
}
__syncthreads ();

int warp_size = 32;

for(int stride = (blockDim.x/2); stride > warp_size; stride = (stride/2)) {	//repeat until stride is 32 (one warp left at this point and no active threads)

if(threadIdx.x < stride) {				// check index range
scratch[threadIdx.x] += scratch[threadIdx.x + stride];
}
__syncthreads ();
}

// manually reduce
if(threadIdx.x <= warp_size)
{
scratch[threadIdx.x] += scratch[threadIdx.x + warp_size];
scratch[threadIdx.x] += scratch[threadIdx.x + warp_size/2];
scratch[threadIdx.x] += scratch[threadIdx.x + warp_size/4];
scratch[threadIdx.x] += scratch[threadIdx.x + warp_size/8];
scratch[threadIdx.x] += scratch[threadIdx.x + warp_size/16];
scratch[threadIdx.x] += scratch[threadIdx.x + 1];
}
__syncthreads ();

if(threadIdx.x == 0) {		// copy back to global array
g_odata[bid] = scratch[0];
}


}