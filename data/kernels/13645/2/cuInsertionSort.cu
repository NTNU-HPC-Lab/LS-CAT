#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

// Variables
int l, i, j;
float *p_dist;
int   *p_ind;
float curr_dist, max_dist;
int   curr_row,  max_row;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){

// Pointer shift, initialization, and max value
p_dist   = dist + xIndex;
p_ind    = ind  + xIndex;
max_dist = p_dist[0];
p_ind[0] = 1;

// Part 1 : sort kth firt elementZ
for (l=1; l<k; l++){
curr_row  = l * dist_pitch;
curr_dist = p_dist[curr_row];
if (curr_dist<max_dist){
i=l-1;
for (int a=0; a<l-1; a++){
if (p_dist[a*dist_pitch]>curr_dist){
i=a;
break;
}
}
for (j=l; j>i; j--){
p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
}
p_dist[i*dist_pitch] = curr_dist;
p_ind[i*ind_pitch]   = l+1;
}
else
p_ind[l*ind_pitch] = l+1;
max_dist = p_dist[curr_row];
}

// Part 2 : insert element in the k-th first lines
max_row = (k-1)*dist_pitch;
for (l=k; l<height; l++){
curr_dist = p_dist[l*dist_pitch];
if (curr_dist<max_dist){
i=k-1;
for (int a=0; a<k-1; a++){
if (p_dist[a*dist_pitch]>curr_dist){
i=a;
break;
}
}
for (j=k-1; j>i; j--){
p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
}
p_dist[i*dist_pitch] = curr_dist;
p_ind[i*ind_pitch]   = l+1;
max_dist             = p_dist[max_row];
}
}
}
}