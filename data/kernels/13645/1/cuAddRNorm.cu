#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec){
unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;
unsigned int xIndex = blockIdx.x * blockDim.x + tx;
unsigned int yIndex = blockIdx.y * blockDim.y + ty;
__shared__ float shared_vec[16];
if (tx==0 && yIndex<height)
shared_vec[ty]=vec[yIndex];
__syncthreads();
if (xIndex<width && yIndex<height)
dist[yIndex*pitch+xIndex]+=shared_vec[ty];
}