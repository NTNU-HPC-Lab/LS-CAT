#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setAllPointsToRemove(bool *d_markers, int number_of_points)
{
int ind=blockIdx.x*blockDim.x+threadIdx.x;
if(ind<number_of_points)
{
d_markers[ind] = false;
}
}