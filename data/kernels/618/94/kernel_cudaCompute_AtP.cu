#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns )
{
int ind=blockIdx.x*blockDim.x+threadIdx.x;
if(ind<rows*columns)
{
int row = ind%rows;
int column = ind/rows;

d_AtP[row + column * rows] = d_A[column + row * columns] * d_P[column];
}
}