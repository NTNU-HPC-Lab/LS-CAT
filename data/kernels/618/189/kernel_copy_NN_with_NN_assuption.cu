#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_copy_NN_with_NN_assuption(double *d_temp_double_mem, int *d_nearest_neighbour_indexes, int number_of_points)
{
int index=blockIdx.x*blockDim.x+threadIdx.x;

if(index < number_of_points)
{
int i = d_nearest_neighbour_indexes[index];
if(i != -1)
{
d_temp_double_mem[index] = 1.0f;
}else
{
d_temp_double_mem[index] = 0.0f;
}
}
}