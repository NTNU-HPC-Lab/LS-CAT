#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;






__global__ void kernel_scanNaiveSumHirizontal(unsigned int *_d_out_integralImage, int _h_width, int _h_height)
{
unsigned int tmp[1024];

int tx = threadIdx.x;



for(int i = 0 ; i < _h_height; i++)
{
tmp[i] = (unsigned int )tex1Dfetch(texIntegralImage, tx + i*_h_width);
}

for(int i = 1; i < _h_height; i++)
{
tmp[i] = tmp[i-1] + tmp[i];
}

for(int i = 0 ; i < _h_height; i++)
{
_d_out_integralImage[tx + i * _h_width] = tmp[i];
}
}