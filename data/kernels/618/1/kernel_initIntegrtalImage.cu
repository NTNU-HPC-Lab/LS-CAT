#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;






__global__ void kernel_initIntegrtalImage(unsigned int *_d_out_integralImage,  int _h_width, int _h_height)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int index = bx + tx*_h_width;
_d_out_integralImage[index] = (unsigned int)0;
}