#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setAllPointsToRemove(int number_of_points, bool *d_markers_out)
{
int ind=blockIdx.x*blockDim.x+threadIdx.x;
if(ind<number_of_points)
{
d_markers_out[ind] = false;
}
}