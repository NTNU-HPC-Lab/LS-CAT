#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_cudaPrepareProjectionIndexes(char *d_v_is_projection, int  *d_nearest_neighbour_indexes,	int number_of_points)
{
int ind=blockIdx.x*blockDim.x+threadIdx.x;

if(ind<number_of_points)
{
if(d_v_is_projection[ind] == 0)
{
d_nearest_neighbour_indexes[ind] = -1;
}else
{
d_nearest_neighbour_indexes[ind] = ind;
}
}
}