#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;






__global__ void kernel_scanNaiveSumVertical(unsigned int *_d_out_integralImage, unsigned char *_d_in_image, int _h_width, int _h_height)
{
unsigned int tmp[1024];

int tx = threadIdx.x;



for(int i = 0 ; i < _h_width; i++)
{
tmp[i] = (unsigned int )tex1Dfetch(texInImage, tx* _h_width + i);
}

for(int i = 1; i < _h_width; i++)
{
tmp[i] = tmp[i-1] + tmp[i];
}

for(int i = 0 ; i < _h_width; i++)
{
_d_out_integralImage[tx* _h_width + i] = tmp[i];
}
}