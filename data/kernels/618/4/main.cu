#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kernel_computeDesctriptorCUDARot.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
bool *_d_isdescriptor = NULL;
hipMalloc(&_d_isdescriptor, XSIZE*YSIZE);
char *_d_vdescriptor = NULL;
hipMalloc(&_d_vdescriptor, XSIZE*YSIZE);
int *_d_keypointsIndexX = NULL;
hipMalloc(&_d_keypointsIndexX, XSIZE*YSIZE);
int *_d_keypointsIndexY = NULL;
hipMalloc(&_d_keypointsIndexY, XSIZE*YSIZE);
int *_d_keypointsRotation = NULL;
hipMalloc(&_d_keypointsRotation, XSIZE*YSIZE);
int _amountofkeypoints = 1;
unsigned int *_d_integralImage = NULL;
hipMalloc(&_d_integralImage, XSIZE*YSIZE);
int _d_width = XSIZE;
int _d_height = YSIZE;
float _scale = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_computeDesctriptorCUDARot<<<gridBlock,threadBlock>>>(_d_isdescriptor,_d_vdescriptor,_d_keypointsIndexX,_d_keypointsIndexY,_d_keypointsRotation,_amountofkeypoints,_d_integralImage,_d_width,_d_height,_scale);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kernel_computeDesctriptorCUDARot<<<gridBlock,threadBlock>>>(_d_isdescriptor,_d_vdescriptor,_d_keypointsIndexX,_d_keypointsIndexY,_d_keypointsRotation,_amountofkeypoints,_d_integralImage,_d_width,_d_height,_scale);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kernel_computeDesctriptorCUDARot<<<gridBlock,threadBlock>>>(_d_isdescriptor,_d_vdescriptor,_d_keypointsIndexX,_d_keypointsIndexY,_d_keypointsRotation,_amountofkeypoints,_d_integralImage,_d_width,_d_height,_scale);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}