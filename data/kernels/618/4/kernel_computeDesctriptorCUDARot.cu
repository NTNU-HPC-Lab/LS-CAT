#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;






__global__ void kernel_computeDesctriptorCUDARot(bool *_d_isdescriptor, char *_d_vdescriptor, int *_d_keypointsIndexX, int *_d_keypointsIndexY, int *_d_keypointsRotation, int _amountofkeypoints, unsigned int *_d_integralImage, int _d_width, int _d_height, float _scale)
{
int bx = blockIdx.x;
int tx = threadIdx.x;

int index = bx + tx*_d_height;
_d_isdescriptor[index] = false;

if(index < _amountofkeypoints)
{
float S[64];
float _X[64];
float _Y[64];
float _Z[64];
float r, phi;
float pi = 3.1415926535f;

for(int i = 0 ; i < 64; i++)
{
r = _scale*pow(2.0f, 2+(i%4));
phi = (float)(i)/4.0f;
_X[i] = (r * cos ((2.0f * pi *phi)/16.0f));
_Y[i] = (r * sin ((2.0f * pi *phi)/16.0f));
_Z[i] = _scale * 8;
}

int _xIndex = _d_keypointsIndexX[index];
int _yIndex = _d_keypointsIndexY[index];
int tau = 4*_d_keypointsRotation[index];

bool check = true;
int index0;
int index1;
int index2;
int index3;

int _h_width = _d_width;
int _h_height = _d_height;

for(int i = 0 ; i < 64; i++)
{
if(int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width) < 0)check = false;
if(int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width) >= _h_width*_h_height)check = false;

if(int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width ) < 0)check = false;
if(int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width ) >= _h_width*_h_height)check = false;

if(int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width )< 0)check = false;
if(int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width )>= _h_width*_h_height)check = false;

if(int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width )< 0)check = false;
if(int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width )>= _h_width*_h_height)check = false;

if(check)
{
index0 = int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width);
index1 = int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width );
index2 = int(_xIndex + floor(_X[i]) + floor(_Z[i]) + (_yIndex + floor(_Y[i]) - floor(_Z[i]))*_h_width );
index3 = int(_xIndex + floor(_X[i]) - floor(_Z[i]) + (_yIndex + floor(_Y[i]) + floor(_Z[i]))*_h_width );

unsigned int a1 = tex1Dfetch(texIntegralImage, index0);
unsigned int a2 = tex1Dfetch(texIntegralImage, index1);
unsigned int a3 = tex1Dfetch(texIntegralImage, index2);
unsigned int a4 = tex1Dfetch(texIntegralImage, index3);

S[i] = float(a1+a2-a3-a4);
}
}

if(check)
{
_d_isdescriptor[index] = true;
bool desc[256];

for(int i = 0; i< 64; i++)
{
int id = (i+tau)%64;
int index0 = (id + 8)%64;
int index1 = (id + 24)%64;
int	index2 = (id + 36)%64;
int index3 = int((4.0f * id/4.0f  + 4.0f + (3 - (id%4))))%64;

if(S[id] < S[index0])
{
desc[i * 4] = true;
}else
{
desc[i * 4] = false;
}

if(S[id] < S[index1])
{
desc[i * 4 + 1] = true;
}else
{
desc[i * 4 + 1] = false;
}

if(S[id] < S[index2])
{
desc[i * 4 + 2] = true;
}else
{
desc[i * 4 + 2] = false;
}

if(S[id] < S[index3])
{
desc[i * 4 + 3] = true;
}else
{
desc[i * 4 + 3] = false;
}
}

for(int i = 0 ; i < 32; i++)
{
char wynik = 0;
for(int j = 0; j < 8; j++)
{
wynik += (desc[i * 8 + j] * (1 << j));
}
_d_vdescriptor[index*32 + i]=wynik;
}
}
}
}