#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  kernel_fill_nn_cuda(unsigned int *d_nn, int *nearest_neighbour_indexes, unsigned int number_nearest_neighbour_indexes)
{
int ind=blockIdx.x*blockDim.x+threadIdx.x;

if(ind < number_nearest_neighbour_indexes)
{
if(nearest_neighbour_indexes[ind] < 0)
{
d_nn[ind] = 0;
}else
{
d_nn[ind] = 1;
}
}
}