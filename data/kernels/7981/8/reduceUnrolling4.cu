#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = (4 * blockIdx.x) * blockDim.x + threadIdx.x;

int *idata = g_idata + (4 * blockIdx.x) * blockDim.x;

if(idx + 3 * blockDim.x < n){
g_idata[idx] += g_idata[idx + blockDim.x];
g_idata[idx] += g_idata[idx + 2 * blockDim.x];
g_idata[idx] += g_idata[idx + 3 * blockDim.x];
}
__syncthreads();

for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
if(tid < stride)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}