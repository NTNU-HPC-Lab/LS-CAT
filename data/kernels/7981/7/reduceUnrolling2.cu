#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = (2 * blockIdx.x) * blockDim.x + threadIdx.x;

int *idata = g_idata + (2 * blockIdx.x) * blockDim.x;

if(idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];
__syncthreads();

for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
if(tid < stride)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}