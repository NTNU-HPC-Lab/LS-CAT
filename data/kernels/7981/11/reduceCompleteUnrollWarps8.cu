#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;

int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

if(idx + 7 * blockDim.x < n){
g_idata[idx] += g_idata[idx + blockDim.x];
g_idata[idx] += g_idata[idx + 2 * blockDim.x];
g_idata[idx] += g_idata[idx + 3 * blockDim.x];
g_idata[idx] += g_idata[idx + 4 * blockDim.x];
g_idata[idx] += g_idata[idx + 5 * blockDim.x];
g_idata[idx] += g_idata[idx + 6 * blockDim.x];
g_idata[idx] += g_idata[idx + 7 * blockDim.x];
}
__syncthreads();

if(blockDim.x >= 1024 && tid < 512)
idata[tid] += idata[tid + 512];
__syncthreads();
if(blockDim.x >= 512 && tid < 256)
idata[tid] += idata[tid + 256];
__syncthreads();
if(blockDim.x >= 256 && tid < 128)
idata[tid] += idata[tid + 128];
__syncthreads();
if(blockDim.x >= 128 && tid < 64)
idata[idx] += idata[tid + 64];
__syncthreads();

if(tid < 32){
volatile int *vmem = idata;
vmem[tid] += vmem[tid + 32];
vmem[tid] += vmem[tid + 16];
vmem[tid] += vmem[tid + 8];
vmem[tid] += vmem[tid + 4];
vmem[tid] += vmem[tid + 2];
vmem[tid] += vmem[tid + 1];
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}