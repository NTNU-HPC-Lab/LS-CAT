#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;

int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

if(idx + 7 * blockDim.x < n){
g_idata[idx] += g_idata[idx + blockDim.x];
g_idata[idx] += g_idata[idx + 2 * blockDim.x];
g_idata[idx] += g_idata[idx + 3 * blockDim.x];
g_idata[idx] += g_idata[idx + 4 * blockDim.x];
g_idata[idx] += g_idata[idx + 5 * blockDim.x];
g_idata[idx] += g_idata[idx + 6 * blockDim.x];
g_idata[idx] += g_idata[idx + 7 * blockDim.x];
}
__syncthreads();

for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
if(tid < stride)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}