#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < N)
C[i] = A[i] + B[i];
}