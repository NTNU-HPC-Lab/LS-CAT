#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmingUp(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

int *idata = g_idata + blockIdx.x * blockDim.x;

if(idx >= n) return ;

for(int stride = 1; stride < blockDim.x; stride <<= 1){
if(tid % (2 * stride) == 0)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}