#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps8(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;

int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

if(idx + 7 * blockDim.x < n){
g_idata[idx] += g_idata[idx + blockDim.x];
g_idata[idx] += g_idata[idx + 2 * blockDim.x];
g_idata[idx] += g_idata[idx + 3 * blockDim.x];
g_idata[idx] += g_idata[idx + 4 * blockDim.x];
g_idata[idx] += g_idata[idx + 5 * blockDim.x];
g_idata[idx] += g_idata[idx + 6 * blockDim.x];
g_idata[idx] += g_idata[idx + 7 * blockDim.x];
}
__syncthreads();

for(int stride = blockDim.x / 2; stride > 32; stride >>= 1){
if(tid < stride)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid < 32){
volatile int *vmem = idata;
vmem[tid] += vmem[tid + 32];
vmem[tid] += vmem[tid + 16];
vmem[tid] += vmem[tid + 8];
vmem[tid] += vmem[tid + 4];
vmem[tid] += vmem[tid + 2];
vmem[tid] += vmem[tid + 1];
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}