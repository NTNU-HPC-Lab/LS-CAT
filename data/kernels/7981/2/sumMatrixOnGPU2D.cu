#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, const int nx, const int ny){
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * nx + ix;
if(ix < nx && iy < ny)
C[idx] = A[idx] + B[idx];
}