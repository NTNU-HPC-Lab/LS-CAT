#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n){
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

int *idata = g_idata + blockIdx.x * blockDim.x;

if(idx >= n) return;

for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
if(tid < stride)
idata[tid] += idata[tid + stride];
__syncthreads();
}

if(tid == 0) g_odata[blockIdx.x] = idata[0];
}