#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tissueGPU4Kernel(int *d_tisspoints, float *d_dtt000, float *d_qtp000, float *d_xt, float *d_rt, int nnt, int step, float diff)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int itp = i/step;
int itp1 = i%step;
int jtp,ixyz,ix,iy,iz,nnt2=2*nnt,istep;
float r = 0.;
if(itp < nnt){
ix = d_tisspoints[itp];
iy = d_tisspoints[itp+nnt];
iz = d_tisspoints[itp+nnt2];
for(jtp=itp1; jtp<nnt; jtp+=step){
ixyz = abs(d_tisspoints[jtp]-ix) + abs(d_tisspoints[jtp+nnt]-iy) + abs(d_tisspoints[jtp+nnt2]-iz);
r -= d_dtt000[ixyz]*d_qtp000[jtp]*d_xt[jtp];
}
r /= diff;
r += d_xt[itp];	//diagonal of matrix has 1s
if(itp1 == 0) d_rt[itp] = r;
}
//The following is apparently needed to assure that d_pt000 is incremented in sequence from the needed threads
for(istep=1; istep<step; istep++){
__syncthreads();
if(itp1 == istep && itp < nnt) d_rt[itp] += r;
}
}