#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
tissueGPU1.cu
GPU kernel to accumulate contributions of tissue source
strengths qt to tissue solute levels pt.
TWS December 2011
Cuda 10.1 Version, August 2019
************************************************************/


__global__ void tissueGPU1Kernel(int *d_tisspoints, float *d_dtt000, float *d_pt000, float *d_qt000, int nnt)
{
int itp = blockDim.x * blockIdx.x + threadIdx.x;
int jtp,ixyz,ix,iy,iz,jx,jy,jz,nnt2=2*nnt;
float p = 0.;
if(itp < nnt){
ix = d_tisspoints[itp];
iy = d_tisspoints[itp+nnt];
iz = d_tisspoints[itp+nnt2];
for(jtp=0; jtp<nnt; jtp++){
jx = d_tisspoints[jtp];
jy = d_tisspoints[jtp+nnt];
jz = d_tisspoints[jtp+nnt2];
ixyz = abs(jx-ix) + abs(jy-iy) + abs(jz-iz);
p += d_qt000[jtp]*d_dtt000[ixyz];
}
d_pt000[itp] = p;
}
}