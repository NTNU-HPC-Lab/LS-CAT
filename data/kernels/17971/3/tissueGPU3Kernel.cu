#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tissueGPU3Kernel(float *d_tissxyz, float *d_vessxyz, float *d_pt000, float *d_qv000, int nnt, int nnv, int is2d, float req, float r2d)
{
int itp = blockDim.x * blockIdx.x + threadIdx.x;
int jvp,nnv2=2*nnv;
float p = 0., xt,yt,zt,x,y,z,dist2,gtv,req2=req*req,r2d2=r2d*r2d;
if(itp < nnt){
xt = d_tissxyz[itp];
yt = d_tissxyz[itp+nnt];
zt = d_tissxyz[itp+nnt*2];
for(jvp=0; jvp<nnv; jvp++){
x = d_vessxyz[jvp] - xt;
y = d_vessxyz[jvp+nnv] - yt;
z = d_vessxyz[jvp+nnv2] - zt;
dist2 = x*x + y*y + z*z;
if(dist2 < req2){
if(is2d) gtv = log(r2d2/req2) + 1. - dist2/req2;
else gtv = (1.5 - 0.5*dist2/req2)/req;
}
else{
if(is2d) gtv = log(r2d2/dist2);
else gtv = 1./sqrt(dist2);
}
p += d_qv000[jvp]*gtv;
}
d_pt000[itp] = p;
}
}