#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tissueGPU3Kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_tissxyz = NULL;
hipMalloc(&d_tissxyz, XSIZE*YSIZE);
float *d_vessxyz = NULL;
hipMalloc(&d_vessxyz, XSIZE*YSIZE);
float *d_pt000 = NULL;
hipMalloc(&d_pt000, XSIZE*YSIZE);
float *d_qv000 = NULL;
hipMalloc(&d_qv000, XSIZE*YSIZE);
int nnt = 1;
int nnv = 1;
int is2d = 1;
float req = 1;
float r2d = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
tissueGPU3Kernel<<<gridBlock,threadBlock>>>(d_tissxyz,d_vessxyz,d_pt000,d_qv000,nnt,nnv,is2d,req,r2d);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
tissueGPU3Kernel<<<gridBlock,threadBlock>>>(d_tissxyz,d_vessxyz,d_pt000,d_qv000,nnt,nnv,is2d,req,r2d);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
tissueGPU3Kernel<<<gridBlock,threadBlock>>>(d_tissxyz,d_vessxyz,d_pt000,d_qv000,nnt,nnv,is2d,req,r2d);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}