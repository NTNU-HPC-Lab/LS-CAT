#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBesselRatioActivationContinuedFraction(float* mat, float* target, float order, int num_terms, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) {
float k = mat[i];
float result = 2 * (order + num_terms) / k;
for(int j = num_terms - 1; j > 0; j--) {
result = 2 * (order + j) / k + 1 / result;
}
target[i] = 1 / result;
}
}