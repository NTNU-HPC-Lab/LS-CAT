#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < numEls; i += numThreads) {
a[i] = a[i] + ((b[i] > 0) ? mult : ((b[i] < 0) ? -mult : 0));
}
}