#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kWriteRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float beta) {
int c = blockIdx.y;
int src_module_id = blockIdx.x;
int dst_module_id = module_id_offset + blockIdx.x;

data += num_images * (src_module_id + c * num_modules_batch);
target += num_images * (dst_module_id + c * num_modules);

for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
target[im] = beta * data[im];
}
}