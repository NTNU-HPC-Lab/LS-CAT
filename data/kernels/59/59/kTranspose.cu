#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTranspose(float *odata, float *idata, int width, int height) {
__shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

// read the matrix tile into shared memory
unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

if((xIndex < width) && (yIndex < height)) {
unsigned int index_in = yIndex * width + xIndex;

block[threadIdx.y][threadIdx.x] = idata[index_in];
}

__syncthreads();

// write the transposed matrix tile to global memory
xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

if((xIndex < height) && (yIndex < width)) {
unsigned int index_out = yIndex * height + xIndex;

odata[index_out] = block[threadIdx.x][threadIdx.y];
}
}