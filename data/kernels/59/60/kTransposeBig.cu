#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTransposeBig(float *odata, float *idata, int height, int width) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
int r, c;
for (unsigned int i = idx; i < width * height; i += numThreads) {
r = i % width;
c = i / width;
odata[i] = idata[height * r + c];
}
}