#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kContract.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *expanded_data = NULL;
hipMalloc(&expanded_data, XSIZE*YSIZE);
float *targets = NULL;
hipMalloc(&targets, XSIZE*YSIZE);
int num_images = 1;
int num_input_channels = 1;
int image_size_y = XSIZE*YSIZE;
int image_size_x = XSIZE*YSIZE;
int num_modules_y = 1;
int num_modules_x = 1;
int kernel_size_y = XSIZE*YSIZE;
int kernel_size_x = XSIZE*YSIZE;
int padding_y = 1;
int padding_x = 1;
int stride_y = 2;
int stride_x = 2;
int num_modules_batch = 2;
int module_id_offset = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kContract<<<gridBlock,threadBlock>>>(expanded_data,targets,num_images,num_input_channels,image_size_y,image_size_x,num_modules_y,num_modules_x,kernel_size_y,kernel_size_x,padding_y,padding_x,stride_y,stride_x,num_modules_batch,module_id_offset);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kContract<<<gridBlock,threadBlock>>>(expanded_data,targets,num_images,num_input_channels,image_size_y,image_size_x,num_modules_y,num_modules_x,kernel_size_y,kernel_size_x,padding_y,padding_x,stride_y,stride_x,num_modules_batch,module_id_offset);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kContract<<<gridBlock,threadBlock>>>(expanded_data,targets,num_images,num_input_channels,image_size_y,image_size_x,num_modules_y,num_modules_x,kernel_size_y,kernel_size_x,padding_y,padding_x,stride_y,stride_x,num_modules_batch,module_id_offset);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}