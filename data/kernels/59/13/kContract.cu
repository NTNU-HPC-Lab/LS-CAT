#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kContract(float *expanded_data, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
int color = blockIdx.y;
int dst_module_id = module_id_offset + blockIdx.x;
int src_module_id = blockIdx.x;

int module_id_x = dst_module_id % num_modules_x;
int module_id_y = dst_module_id / num_modules_x;
int startX = module_id_x * stride_x + padding_x;
int startY = module_id_y * stride_y + padding_y;
int Y, X;
long target_id, source_id;
targets += num_images * image_size_x * image_size_y * color;
expanded_data  += num_images * (src_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
for (int y = 0; y < kernel_size_y; y++) {
Y = startY + y;
for (int x = 0; x < kernel_size_x; x++) {
X = startX + x;
source_id = num_images * num_modules_batch * (x + kernel_size_x * y);
target_id = num_images * (X + image_size_x * Y);
if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
// do nothing.
} else {
for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
atomicAdd(&targets[target_id + im], expanded_data[source_id + im]);
__syncthreads();
}
}
}
}
}