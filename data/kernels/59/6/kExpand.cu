#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExpand(float *images, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
int color = blockIdx.y;
int src_module_id = module_id_offset + blockIdx.x;
int dst_module_id = blockIdx.x;

int module_id_x = src_module_id % num_modules_x;
int module_id_y = src_module_id / num_modules_x;
int startX = module_id_x * stride_x + padding_x;
int startY = module_id_y * stride_y + padding_y;
int Y, X;
long target_id, source_id;
images += num_images * image_size_x * image_size_y * color;
targets += num_images * (dst_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
for (int y = 0; y < kernel_size_y; y++) {
Y = startY + y;
for (int x = 0; x < kernel_size_x; x++) {
X = startX + x;
target_id = num_images * num_modules_batch * (x + kernel_size_x * y);
source_id = num_images * (X + image_size_x * Y);
if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
targets[target_id + im] = 0;
}
} else {
for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
targets[target_id + im] = images[source_id + im];
}
}
__syncthreads();
}
}
}