#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg){
const int row = gridDim.x * blockIdx.y + blockIdx.x;
const int column = threadIdx.x;
if (row < height && column < target_width) {
float cur_sum = 0.0;
unsigned int count = 0;
for (unsigned int i = 0; i < mat_width; i ++) {
count += ((int)indices[i] == column) ? 1 : 0 ;
cur_sum += ((int)indices[i] == column) ? mat[row + i * height] : 0 ;
}
target[row + height * column] = mult * cur_sum / ((avg == 1 && count > 0) ? count : 1);
}
}