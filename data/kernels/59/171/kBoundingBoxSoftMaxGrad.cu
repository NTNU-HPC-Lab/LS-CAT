#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBoundingBoxSoftMaxGrad( float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
const unsigned int len = width * height * depth * size;
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
int ind, image_id, source_depth, x1, y1, x2, y2, start,
end, src_image_id, num_bboxes, num_bboxes_of_this_depth, box_id, inside;
float source_x, source_y;
for (unsigned int i = idx; i < len; i += numThreads) {
ind = i;
image_id = ind % size; ind /= size;
source_x = scale_width * (ind % width); ind /= width;
source_y = scale_height * (ind % height); ind /= height;
source_depth = ind % depth;
src_image_id = (int)indices[image_id];

start = seg[src_image_id];
end = seg[src_image_id + 1];
num_bboxes = 0;
num_bboxes_of_this_depth = 0;
for (box_id = start; box_id < end; box_id++) {
x1 = bbox[box_id << 2] - width_offset[image_id];
y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1:0;
num_bboxes += inside;
num_bboxes_of_this_depth += (inside == 1 && label[box_id] == source_depth) ? 1: 0;
}
grad[i] = mat[i] - ((num_bboxes > 0) ? ((float)num_bboxes_of_this_depth / num_bboxes) : (source_depth == 0 ? 1:0));
}
}