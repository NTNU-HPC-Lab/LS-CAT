#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
__shared__ int sourceRowIndices[32];
const int startTargetRowI = blockIdx.x * 32;
const int tid = threadIdx.x;
const int localNRowIs = min(32, nRowIs-startTargetRowI);

// cooperatively load 32 row indices
if (tid < localNRowIs){
sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
if (sourceRowIndices[tid]<0)
sourceRowIndices[tid] += nSourceRows;
if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
sourceRowIndices[tid] = -1;
}
__syncthreads();

// copy 32 rows
for (int i=0; i<localNRowIs; i++){
const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
for (int colI=tid; colI<nCols; colI+=32)
target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
}
}