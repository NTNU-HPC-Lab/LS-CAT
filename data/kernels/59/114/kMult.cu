#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls, float scale_targets) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
if (scale_targets == 0) {
for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = a[i] * b[i];
}
} else {
for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = scale_targets * dest[i] + a[i] * b[i];
}
}
}