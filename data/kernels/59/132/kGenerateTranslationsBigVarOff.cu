#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
const unsigned int idx = threadIdx.x;
const unsigned int numThreads = blockDim.x;

int target_x, target_y;
int pad = (source_w - target_w)/2;
int target_tile_size = target_w * target_w;
int source_tile_size = source_w * source_w;

int off_x = off_x_arr[blockIdx.x];
int off_y = off_y_arr[blockIdx.x];
int target_off = blockIdx.x * target_tile_size;
int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
target_x = target_ind / target_w;
target_y = target_ind - target_x * target_w;

for (unsigned int ch = 0; ch < num_channels; ch += 1) {
target[num_channels*(target_off + target_x * target_w + target_y) + ch] = source[num_channels*(source_off + target_x * source_w + target_y) + ch];
}
}
}