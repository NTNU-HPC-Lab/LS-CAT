#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultByRowVectorScale(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scale_targets) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = scale_targets * tgtMat[i] + mat[i] * vec[i / height];
}
}