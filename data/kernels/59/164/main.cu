#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kExtractPatches3.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *images = NULL;
hipMalloc(&images, XSIZE*YSIZE);
float *patches = NULL;
hipMalloc(&patches, XSIZE*YSIZE);
float *width_offset = NULL;
hipMalloc(&width_offset, XSIZE*YSIZE);
float *height_offset = NULL;
hipMalloc(&height_offset, XSIZE*YSIZE);
float *flip = NULL;
hipMalloc(&flip, XSIZE*YSIZE);
int num_images = 1;
int img_width = XSIZE;
int img_height = YSIZE;
int patch_width = XSIZE;
int patch_height = YSIZE;
int num_colors = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kExtractPatches3<<<gridBlock,threadBlock>>>(images,patches,width_offset,height_offset,flip,num_images,img_width,img_height,patch_width,patch_height,num_colors);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kExtractPatches3<<<gridBlock,threadBlock>>>(images,patches,width_offset,height_offset,flip,num_images,img_width,img_height,patch_width,patch_height,num_colors);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kExtractPatches3<<<gridBlock,threadBlock>>>(images,patches,width_offset,height_offset,flip,num_images,img_width,img_height,patch_width,patch_height,num_colors);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}