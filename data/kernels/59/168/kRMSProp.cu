#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRMSProp(float *history, float *grad, float factor, int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) {
history[i] = sqrt(factor * history[i] * history[i] + (1-factor) * grad[i] * grad[i]);
}
}