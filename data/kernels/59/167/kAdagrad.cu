#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdagrad(float *history, float *grad, float delta, int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < len; i += numThreads) {
float curr_norm = history[i] - delta;
history[i] = delta + sqrt(curr_norm * curr_norm + grad[i] * grad[i]);
}
}