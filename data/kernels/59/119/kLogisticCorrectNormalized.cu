#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogisticCorrectNormalized(float* mat, float* targets, float* out, unsigned int height, unsigned int width) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < height) {
float correct = 0;
float total = 0;
float p, t;
for (int i = idx; i < width * height; i += height) {
p = mat[i];
t = targets[i];
correct += (t < 0) ? 0 : (((t >= 0.5 && p >= 0.5) || (t < 0.5 && p < 0.5)) ? 1: 0);
total += (t < 0) ? 0 : 1;
__syncthreads();
}
out[idx] = (total > 0) ? (correct / total) : 0;
}
}