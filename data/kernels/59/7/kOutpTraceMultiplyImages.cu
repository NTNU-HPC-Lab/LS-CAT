#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kOutpTraceMultiplyImages(float *expanded_images, float *expanded_derivs, int num_images, int num_channels, int num_modules_batch, int kernel_size){
int color = blockIdx.y;
int module_id = blockIdx.x;

expanded_images += num_images * num_modules_batch * kernel_size * color;
expanded_images += num_images * module_id;
expanded_derivs += num_images * num_modules_batch * color;
expanded_derivs += num_images * module_id;

for (int kpos = 0; kpos < kernel_size; kpos++) {
for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
int image_idx = im + num_images * num_modules_batch * kpos;
int deriv_idx = im;
expanded_images[image_idx] *= expanded_derivs[deriv_idx];
}
__syncthreads();
}

}