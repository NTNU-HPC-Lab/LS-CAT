#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
__shared__ int targetRowIndices[32];
const int startSourceRowI = blockIdx.x * 32;
const int tid = threadIdx.x;
const int localNRowIs = min(32, nRowIs-startSourceRowI);

// cooperatively load 32 row indices
if (tid < localNRowIs){
targetRowIndices[tid] = int(indices[startSourceRowI + tid]);
if (targetRowIndices[tid]<0)
targetRowIndices[tid] += nTargetRows;
if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nTargetRows)
targetRowIndices[tid] = -1;
}
__syncthreads();

// copy 32 rows
for (int i=0; i<localNRowIs; i++){
const int sourceRowI = startSourceRowI + i, targetRowI = targetRowIndices[i];
for (int colI=tid; colI<nCols; colI+=32)
target[targetRowI * nCols + colI] = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
}
}