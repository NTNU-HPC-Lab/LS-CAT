#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static float flow_absolute_residual(float x, float y, float ux, float uy, float d, float fx, float fy, float T0, float T1, float T2, float R0, float R1, float R2) {
float rx = -ux + fx * R1 - y * R2 + ((x * x) * R1) / fx + d * fx * T0 -
d * x * T2 - (x * y * R0) / fx;
float ry = -uy - fy * R0 + x * R2 - d * y * T2 - ((y * y) * R0) / fy +
d * fy * T1 + (x * y * R1) / fy;

return sqrtf(rx * rx + ry * ry);
}
__global__ void flow_absolute_residual_scalable_GPU( float *d_abs_res, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, const unsigned int *d_valid_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, int n_valid_flow_Zbuffer, const int *d_offset_ind, const int *d_segment_translation_table, float w_flow, float w_ar_flow, const float *d_dTR) {

int ind = blockDim.x * blockIdx.x + threadIdx.x;

if (ind < n_valid_flow_Zbuffer) {

// determine current segment
int segment = d_segment_translation_table[d_valid_flow_Zbuffer[ind]];

// fetch flow and Zbuffer from global memory
float2 u = d_flow_compact[ind];
float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[ind]);

// compute coordinates
int pixel_ind = d_ind_flow_Zbuffer[ind];
bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = x - ox;
y = y - oy;

// compute absolute residual
// here the weights will be introduced
float w = is_ar_flow ? w_ar_flow : w_flow;
int ind_out = ind + d_offset_ind[segment];
int s6 = segment * 6;
d_abs_res[ind_out] =
w * flow_absolute_residual(x, y, u.x, u.y, disp, fx, fy, d_dTR[s6],
d_dTR[s6 + 1], d_dTR[s6 + 2], d_dTR[s6 + 3],
d_dTR[s6 + 4], d_dTR[s6 + 5]);
}
}