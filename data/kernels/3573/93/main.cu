#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "flow_absolute_residual_scalable_GPU.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_abs_res = NULL;
hipMalloc(&d_abs_res, XSIZE*YSIZE);
const float2 *d_flow_compact = NULL;
hipMalloc(&d_flow_compact, XSIZE*YSIZE);
const float *d_Zbuffer_flow_compact = NULL;
hipMalloc(&d_Zbuffer_flow_compact, XSIZE*YSIZE);
const int *d_ind_flow_Zbuffer = NULL;
hipMalloc(&d_ind_flow_Zbuffer, XSIZE*YSIZE);
const unsigned int *d_valid_flow_Zbuffer = NULL;
hipMalloc(&d_valid_flow_Zbuffer, XSIZE*YSIZE);
float fx = 1;
float fy = 1;
float ox = 1;
float oy = 1;
int n_rows = 1;
int n_cols = 1;
int n_valid_flow_Zbuffer = 1;
const int *d_offset_ind = NULL;
hipMalloc(&d_offset_ind, XSIZE*YSIZE);
const int *d_segment_translation_table = NULL;
hipMalloc(&d_segment_translation_table, XSIZE*YSIZE);
float w_flow = 1;
float w_ar_flow = 1;
const float *d_dTR = NULL;
hipMalloc(&d_dTR, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
flow_absolute_residual_scalable_GPU<<<gridBlock,threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
flow_absolute_residual_scalable_GPU<<<gridBlock,threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
flow_absolute_residual_scalable_GPU<<<gridBlock,threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}