#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorDistDiff_kernel(uchar4 *out_image, const float *disparity, int disparity_pitch, const float *disparity_prior, int width, int height, float f, float b, float ox, float oy, float dist_thres) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < width && y < height) {

int ind = y * width + x;
uchar4 temp = out_image[ind];
float disp = *((float *)((char *)disparity + y * disparity_pitch) + x);
float disp_model = disparity_prior[ind];

// 3D reconstruct and measure Euclidian distance
float xt = __fdividef((x - ox), f);
float yt = -__fdividef((y - oy), f); // coord. transform

float Zm = -(f * b) / disp_model;
float Xm = xt * Zm;
float Ym = yt * Zm;

float Zd = -(f * b) / disp;
float Xd = xt * Zd;
float Yd = yt * Zd;

float d_md = sqrtf((Xm - Xd) * (Xm - Xd) + (Ym - Yd) * (Ym - Yd) +
(Zm - Zd) * (Zm - Zd));

bool color = (d_md > dist_thres) | (isfinite(disp) & ~isfinite(disp_model));

if (color) { // color
temp.x *= 0.5f;
temp.y *= 0.5f;
}

out_image[ind] = temp;
}
}