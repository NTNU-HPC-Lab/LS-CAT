#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void median_reduce_shuffle_gpu(const float *d_in, float *d_out, float *d_random_numbers, int n_in) {

/**************/
/* initialize */
/**************/

// compute indices

int t_ind = threadIdx.x;
int g_ind = blockIdx.x * MED_BLOCK_SIZE + t_ind;

// allocate shared memory

__shared__ float DATA[MED_BLOCK_SIZE];

/**************/
/* load stage */
/**************/

int sample_ind = floorf(d_random_numbers[g_ind] * (float)n_in);
DATA[t_ind] = d_in[sample_ind];

__syncthreads();

/*******************/
/* reduction stage */
/*******************/

for (int s = 1; s < MED_BLOCK_SIZE; s *= 3) {

int index = 3 * s * t_ind;

if (index < MED_BLOCK_SIZE) {

// fetch three values
float value1 = DATA[index];
float value2 = DATA[index + s];
float value3 = DATA[index + 2 * s];

// extract the middle value (median)
float smallest = fminf(value1, value2);
value2 = fmaxf(value1, value2);
value1 = smallest;

value3 = fmaxf(value1, value3);
value2 = fminf(value2, value3);

DATA[index] = value2;
}

__syncthreads();
}

/***************/
/* write stage */
/***************/

// write this block's approx median (first element)

if (t_ind == 0) {
d_out[blockIdx.x] = DATA[0];
}
}