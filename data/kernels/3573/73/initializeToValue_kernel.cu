#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeToValue_kernel(unsigned int *data, unsigned int value, int width, int height) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < width && y < height) {
data[y * width + x] = value;
}
}