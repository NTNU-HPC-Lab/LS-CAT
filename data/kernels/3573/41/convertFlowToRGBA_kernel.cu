#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertFlowToRGBA_kernel(uchar4 *d_flowx_out, uchar4 *d_flowy_out, const float *d_flowx_in, const float *d_flowy_in, int width, int height, float lowerLim, float upperLim, float minMag) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 tempx, tempy;
if (x < width && y < height) {
float ux = d_flowx_in[__mul24(y, width) + x];
float uy = d_flowy_in[__mul24(y, width) + x];

float mag = sqrtf(ux * ux + uy * uy);

// first draw unmatched pixels in white
if (!isfinite(ux) || (mag < minMag)) {

tempx.x = 255;
tempx.y = 255;
tempx.z = 255;
tempx.w = 255;
tempy.x = 255;
tempy.y = 255;
tempy.z = 255;
tempy.w = 255;

} else {

// rescale value from [lowerLim,upperLim] to [0,1]
ux -= lowerLim;
ux /= (upperLim - lowerLim);

float r = 1.0f;
float g = 1.0f;
float b = 1.0f;
if (ux < 0.25f) {
r = 0;
g = 4.0f * ux;
} else if (ux < 0.5f) {
r = 0;
b = 1.0 + 4.0f * (0.25f - ux);
} else if (ux < 0.75f) {
r = 4.0f * (ux - 0.5f);
b = 0;
} else {
g = 1.0f + 4.0f * (0.75f - ux);
b = 0;
}
tempx.x = 255.0 * r;
tempx.y = 255.0 * g;
tempx.z = 255.0 * b;
tempx.w = 255;

uy -= lowerLim;
uy /= (upperLim - lowerLim);

r = 1.0f;
g = 1.0f;
b = 1.0f;
if (uy < 0.25f) {
r = 0;
g = 4.0f * uy;
} else if (uy < 0.5f) {
r = 0;
b = 1.0 + 4.0f * (0.25f - uy);
} else if (uy < 0.75f) {
r = 4.0f * (uy - 0.5f);
b = 0;
} else {
g = 1.0f + 4.0f * (0.75f - uy);
b = 0;
}
tempy.x = 255.0 * r;
tempy.y = 255.0 * g;
tempy.z = 255.0 * b;
tempy.w = 255;
}

d_flowx_out[__mul24(y, width) + x] = tempx;
d_flowy_out[__mul24(y, width) + x] = tempy;
}
}