#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample_ind_and_labels_GPU(int *d_ind_sub, const int *d_ind, unsigned int *d_label_sub, const unsigned int *d_label, int n_out, float inv_sub_factor) {

unsigned int ind_out = blockIdx.x * blockDim.x + threadIdx.x;

if (ind_out < n_out) {

int ind_in = (int)floorf((float)(ind_out) * inv_sub_factor);
d_ind_sub[ind_out] = d_ind[ind_in];
d_label_sub[ind_out] = d_label[ind_in];
}
}