#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "normal_eqs_flow_multicam_GPU.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_CO = NULL;
hipMalloc(&d_CO, XSIZE*YSIZE);
float2 *d_flow_compact = NULL;
hipMalloc(&d_flow_compact, XSIZE*YSIZE);
float *d_Zbuffer_flow_compact = NULL;
hipMalloc(&d_Zbuffer_flow_compact, XSIZE*YSIZE);
int *d_ind_flow_Zbuffer = NULL;
hipMalloc(&d_ind_flow_Zbuffer, XSIZE*YSIZE);
const float *d_focal_length = NULL;
hipMalloc(&d_focal_length, XSIZE*YSIZE);
const float *d_nodal_point_x = NULL;
hipMalloc(&d_nodal_point_x, XSIZE*YSIZE);
const float *d_nodal_point_y = NULL;
hipMalloc(&d_nodal_point_y, XSIZE*YSIZE);
const int *d_n_rows = NULL;
hipMalloc(&d_n_rows, XSIZE*YSIZE);
const int *d_n_cols = NULL;
hipMalloc(&d_n_cols, XSIZE*YSIZE);
const int *d_n_values_flow = NULL;
hipMalloc(&d_n_values_flow, XSIZE*YSIZE);
const int *d_start_ind_flow = NULL;
hipMalloc(&d_start_ind_flow, XSIZE*YSIZE);
const int *d_pixel_ind_offset = NULL;
hipMalloc(&d_pixel_ind_offset, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_flow_multicam_GPU<<<gridBlock,threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
normal_eqs_flow_multicam_GPU<<<gridBlock,threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
normal_eqs_flow_multicam_GPU<<<gridBlock,threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}