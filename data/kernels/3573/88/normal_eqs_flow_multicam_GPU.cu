#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_flow_multicam_GPU( float *d_CO, float2 *d_flow_compact, float *d_Zbuffer_flow_compact, int *d_ind_flow_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const int *d_n_rows, const int *d_n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const int *d_pixel_ind_offset) {
int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be
// multiple of blocksize

int n_flow = d_n_values_flow[blockIdx.y];
int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
int start_ind = d_start_ind_flow[blockIdx.y];

float f = d_focal_length[blockIdx.y];
float ox = d_nodal_point_x[blockIdx.y];
float oy = d_nodal_point_y[blockIdx.y];
int n_rows = d_n_rows[blockIdx.y];
int n_cols = d_n_cols[blockIdx.y];
int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_flow) { // is this a valid sample?

// fetch flow and Zbuffer from global memory
float2 u = d_flow_compact[in_ind + start_ind];
float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

// compute coordinates
int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind] - pixel_ind_offset;
bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = x - ox;
y = y - oy;

// flip y axis
y = -y;
u.y = -u.y;

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix
A0 += (disp * disp * f * f);
A1 += (-disp * disp * x * f);
A2 += (-disp * x * y);
A3 += (disp * f * f + disp * x * x);
A4 += (-disp * y * f);
A5 += (-disp * disp * y * f);
A6 += (-disp * f * f - disp * y * y);
A7 += (disp * x * f);
A8 += (disp * disp * x * x + disp * disp * y * y);
A9 += (disp * x * x * y / f + disp * y * f + disp * y * y * y / f);
A10 += (-disp * x * f - disp * x * x * x / f - disp * x * y * y / f);
A11 += (x * x * y * y / (f * f) + f * f + 2.0f * y * y +
y * y * y * y / (f * f));
A12 +=
(-2.0f * x * y - x * x * x * y / (f * f) - x * y * y * y / (f * f));
A13 += (-x * f);
A14 += (f * f + 2.0f * x * x + x * x * x * x / (f * f) +
x * x * y * y / (f * f));
A15 += (-y * f);
A16 += (x * x + y * y);

// B-vector

A17 += (disp * u.x * f);
A18 += (disp * u.y * f);
A19 += (-disp * x * u.x - disp * y * u.y);
A20 += (-x * y * u.x / f - u.y * f - u.y * y * y / f);
A21 += (u.x * f + x * x * u.x / f + x * y * u.y / f);
A22 += (-y * u.x + x * u.y);
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

d_CO[out_ind] = A0;
d_CO[out_ind + n_val_accum] = A1;
d_CO[out_ind + 2 * n_val_accum] = A2;
d_CO[out_ind + 3 * n_val_accum] = A3;
d_CO[out_ind + 4 * n_val_accum] = A4;
d_CO[out_ind + 5 * n_val_accum] = A5;
d_CO[out_ind + 6 * n_val_accum] = A6;
d_CO[out_ind + 7 * n_val_accum] = A7;
d_CO[out_ind + 8 * n_val_accum] = A8;
d_CO[out_ind + 9 * n_val_accum] = A9;
d_CO[out_ind + 10 * n_val_accum] = A10;
d_CO[out_ind + 11 * n_val_accum] = A11;
d_CO[out_ind + 12 * n_val_accum] = A12;
d_CO[out_ind + 13 * n_val_accum] = A13;
d_CO[out_ind + 14 * n_val_accum] = A14;
d_CO[out_ind + 15 * n_val_accum] = A15;
d_CO[out_ind + 16 * n_val_accum] = A16;
d_CO[out_ind + 17 * n_val_accum] = A17;
d_CO[out_ind + 18 * n_val_accum] = A18;
d_CO[out_ind + 19 * n_val_accum] = A19;
d_CO[out_ind + 20 * n_val_accum] = A20;
d_CO[out_ind + 21 * n_val_accum] = A21;
d_CO[out_ind + 22 * n_val_accum] = A22;
}