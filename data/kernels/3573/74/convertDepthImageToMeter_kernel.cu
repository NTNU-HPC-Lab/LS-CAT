#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertDepthImageToMeter_kernel(float *d_depth_image_meter, const unsigned int *d_depth_image_millimeter, int n_rows, int n_cols) {

const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n_cols && y < n_rows) {
int ind = y * n_cols + x;
unsigned int depth = d_depth_image_millimeter[ind];
d_depth_image_meter[ind] =
(depth == 4294967295) ? nanf("") : (float)depth / 1000.0f;
}
}