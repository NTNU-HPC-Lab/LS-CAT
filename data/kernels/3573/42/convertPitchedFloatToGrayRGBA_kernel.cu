#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPitchedFloatToGrayRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

uchar4 temp;

if (x < width && y < height) {
//    float val = in_image[__mul24(y,pitch)+x];
float val = *((float *)((char *)in_image + y * pitch) + x);

// rescale value from [lowerLim,upperLim] to [0,255]
val -= lowerLim;
val /= (upperLim - lowerLim);
val *= 255.0;

temp.x = val;
temp.y = val;
temp.z = val;
temp.w = 255;

out_image[__mul24(y, width) + x] = temp;
}
}