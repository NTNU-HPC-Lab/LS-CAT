#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static float disp_absolute_residual(float Xd, float Yd, float Zd, float Xm, float Ym, float Zm, float nx, float ny, float nz, float T0, float T1, float T2, float R0, float R1, float R2, float fx, float b) {
float r = -Xd * nx + Xm * nx - Yd * ny + Ym * ny - Zd * nz + Zm * nz +
nx * T0 + ny * T1 + nz * T2 + Xm * ny * R2 - Xm * nz * R1 -
Ym * nx * R2 + Ym * nz * R0 + Zm * nx * R1 - Zm * ny * R0;

// weight to convert distance units to pixels
r *= fx * b / (Zm * Zm);

return fabsf(r);
}
__global__ void disp_absolute_residual_scalable_GPU( float *d_abs_res, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, const unsigned int *d_valid_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, int n_valid_disparity_Zbuffer, const int *d_offset_ind, const int *d_segment_translation_table, float w_disp, const float *d_dTR) {

int ind = blockDim.x * blockIdx.x + threadIdx.x;

if (ind < n_valid_disparity_Zbuffer) {

// determine current segment
int segment = d_segment_translation_table[d_valid_disparity_Zbuffer[ind]];

// fetch disparity, Zbuffer and normal from global memory
float disp = d_disparity_compact[ind];
float4 tmp = d_Zbuffer_normals_compact[ind];
float Zbuffer = tmp.x;
float nx = tmp.y;
float ny = tmp.z;
float nz = tmp.w;

// compute coordinates
int pixel_ind = d_ind_disparity_Zbuffer[ind];

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = __fdividef((x - ox), fx);
y = __fdividef((y - oy), fy);

// reconstruct 3D point from disparity
float Zd = -(fx * b) / disp; // arbitrary use of fx for now
float Xd = x * Zd;
float Yd = y * Zd;

// reconstruct 3D point from model
float Zm = Zbuffer;
float Xm = x * Zm;
float Ym = y * Zm;

// compute absolute residual (weighted by disparity vs flow importance)
int ind_out = ind + d_offset_ind[segment];
int s6 = segment * 6;
d_abs_res[ind_out] =
w_disp * disp_absolute_residual(Xd, Yd, Zd, Xm, Ym, Zm, nx, ny, nz,
d_dTR[s6], d_dTR[s6 + 1], d_dTR[s6 + 2],
d_dTR[s6 + 3], d_dTR[s6 + 4],
d_dTR[s6 + 5], fx, b);
}
}