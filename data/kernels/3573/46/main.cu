#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "convert2DVectorToAngleMagnitude_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
uchar4 *d_angle_image = NULL;
hipMalloc(&d_angle_image, XSIZE*YSIZE);
uchar4 *d_magnitude_image = NULL;
hipMalloc(&d_magnitude_image, XSIZE*YSIZE);
float *d_vector_X = NULL;
hipMalloc(&d_vector_X, XSIZE*YSIZE);
float *d_vector_Y = NULL;
hipMalloc(&d_vector_Y, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
float lower_ang = 1;
float upper_ang = 1;
float lower_mag = 1;
float upper_mag = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
convert2DVectorToAngleMagnitude_kernel<<<gridBlock,threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
convert2DVectorToAngleMagnitude_kernel<<<gridBlock,threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
convert2DVectorToAngleMagnitude_kernel<<<gridBlock,threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}