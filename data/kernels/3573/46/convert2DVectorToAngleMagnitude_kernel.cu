#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convert2DVectorToAngleMagnitude_kernel( uchar4 *d_angle_image, uchar4 *d_magnitude_image, float *d_vector_X, float *d_vector_Y, int width, int height, float lower_ang, float upper_ang, float lower_mag, float upper_mag) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 temp_angle, temp_magnitude;

if (x < width && y < height) {
float vector_X = d_vector_X[__mul24(y, width) + x];
float vector_Y = d_vector_Y[__mul24(y, width) + x];

// compute angle and magnitude
float angle = atan2f(vector_Y, vector_X);
float magnitude = vector_X * vector_X + vector_Y * vector_Y;
magnitude = sqrtf(magnitude);

// first draw unmatched pixels in white
if (!isfinite(magnitude)) {
temp_angle.x = 255;
temp_angle.y = 255;
temp_angle.z = 255;
temp_angle.w = 255;
temp_magnitude.x = 255;
temp_magnitude.y = 255;
temp_magnitude.z = 255;
temp_magnitude.w = 255;
} else {
// rescale angle and magnitude from [lower,upper] to [0,1] and convert to
// RGBA jet colorspace

angle -= lower_ang;
angle /= (upper_ang - lower_ang);

float r = 1.0f;
float g = 1.0f;
float b = 1.0f;

if (angle < 0.25f) {
r = 0;
g = 4.0f * angle;
} else if (angle < 0.5f) {
r = 0;
b = 1.0 + 4.0f * (0.25f - angle);
} else if (angle < 0.75f) {
r = 4.0f * (angle - 0.5f);
b = 0;
} else {
g = 1.0f + 4.0f * (0.75f - angle);
b = 0;
}

temp_angle.x = 255.0 * r;
temp_angle.y = 255.0 * g;
temp_angle.z = 255.0 * b;
temp_angle.w = 255;

magnitude -= lower_mag;
magnitude /= (upper_mag - lower_mag);

r = 1.0f;
g = 1.0f;
b = 1.0f;

if (magnitude < 0.25f) {
r = 0;
g = 4.0f * magnitude;
} else if (magnitude < 0.5f) {
r = 0;
b = 1.0 + 4.0f * (0.25f - magnitude);
} else if (magnitude < 0.75f) {
r = 4.0f * (magnitude - 0.5f);
b = 0;
} else {
g = 1.0f + 4.0f * (0.75f - magnitude);
b = 0;
}

temp_magnitude.x = 255.0 * r;
temp_magnitude.y = 255.0 * g;
temp_magnitude.z = 255.0 * b;
temp_magnitude.w = 255;
}

d_angle_image[__mul24(y, width) + x] = temp_angle;
d_magnitude_image[__mul24(y, width) + x] = temp_magnitude;
}
}