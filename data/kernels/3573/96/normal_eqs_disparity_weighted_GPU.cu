#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static float disp_absolute_residual(float Xd, float Yd, float Zd, float Xm, float Ym, float Zm, float nx, float ny, float nz, float T0, float T1, float T2, float R0, float R1, float R2, float fx, float b) {
float r = -Xd * nx + Xm * nx - Yd * ny + Ym * ny - Zd * nz + Zm * nz +
nx * T0 + ny * T1 + nz * T2 + Xm * ny * R2 - Xm * nz * R1 -
Ym * nx * R2 + Ym * nz * R0 + Zm * nx * R1 - Zm * ny * R0;

// weight to convert distance units to pixels
r *= fx * b / (Zm * Zm);

return fabsf(r);
}
__global__ void normal_eqs_disparity_weighted_GPU( float *d_CD, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, const float *d_abs_res_scales, float w_disp, const float *d_dTR) {

int n_val_accum =
gridDim.x * blockDim.x; // n_val_accum may not be multiple of blocksize

int n_disparity = d_n_values_disparity[blockIdx.y];
int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
int start_ind = d_start_ind_disparity[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f, A23 = 0.0f,
A24 = 0.0f, A25 = 0.0f, A26 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_disparity) { // is this a valid sample?

// fetch disparity, Zbuffer and normal from global memory
float disp = d_disparity_compact[in_ind + start_ind];
float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
float Zbuffer = tmp.x;
float nx = tmp.y;
float ny = tmp.z;
float nz = tmp.w;

// compute coordinates
int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind];

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = __fdividef((x - ox), fx);
y = __fdividef((y - oy), fy);

// reconstruct 3D point from disparity

float Zd = -(fx * b) / disp; // arbitrary use of fx
float Xd = x * Zd;
float Yd = y * Zd;

// reconstruct 3D point from model

float Zm = Zbuffer;
float Xm = x * Zm;
float Ym = y * Zm;

// determine M-estimation weight
// disparity residual weighed by rel. importance disp vs flow
int s6 = blockIdx.y * 6;
float w = w_disp * disp_absolute_residual(
Xd, Yd, Zd, Xm, Ym, Zm, nx, ny, nz, d_dTR[s6],
d_dTR[s6 + 1], d_dTR[s6 + 2], d_dTR[s6 + 3],
d_dTR[s6 + 4], d_dTR[s6 + 5], fx, b);
w /= d_abs_res_scales[blockIdx.y];
w = (w > 1) ? 0 : (1.0f - 2.0f * w * w + w * w * w * w);

// multiply m estimation weight with distance->pixel conversion weight
// (squared)
w *= (fx * fx * b * b) / (Zm * Zm * Zm * Zm);

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix

A0 += w * (nx * nx);
A1 += w * (nx * ny);
A2 += w * (nx * nz);
A3 += w * (Ym * nx * nz - Zm * nx * ny);
A4 += w * (Zm * (nx * nx) - Xm * nx * nz);
A5 += w * (-Ym * (nx * nx) + Xm * nx * ny);

A6 += w * (ny * ny);
A7 += w * (ny * nz);
A8 += w * (-Zm * (ny * ny) + Ym * ny * nz);
A9 += w * (-Xm * ny * nz + Zm * nx * ny);
A10 += w * (Xm * (ny * ny) - Ym * nx * ny);

A11 += w * (nz * nz);
A12 += w * (Ym * (nz * nz) - Zm * ny * nz);
A13 += w * (-Xm * (nz * nz) + Zm * nx * nz);
A14 += w * (Xm * ny * nz - Ym * nx * nz);

A15 += w * ((Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) -
Ym * Zm * ny * nz * 2.0f);
A16 += w * (-Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny +
Xm * Zm * ny * nz + Ym * Zm * nx * nz);
A17 += w * (-Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz +
Xm * Ym * ny * nz + Ym * Zm * nx * ny);

A18 += w * ((Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) -
Xm * Zm * nx * nz * 2.0f);
A19 += w * (-Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz +
Xm * Ym * nx * nz + Xm * Zm * nx * ny);

A20 += w * ((Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) -
Xm * Ym * nx * ny * 2.0f);

// B-vector

A21 += w * (Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny -
Ym * nx * ny + Zd * nx * nz - Zm * nx * nz);
A22 += w * (Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny -
Xm * nx * ny + Zd * ny * nz - Zm * ny * nz);
A23 += w * (Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz -
Xm * nx * nz + Yd * ny * nz - Ym * ny * nz);
A24 += w *
(-Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) -
Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny +
Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz);
A25 +=
w * (Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) +
Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz +
Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz);
A26 += w *
(-Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) -
Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz -
Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz);
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

w_disp *= w_disp; // weight relative to flow

d_CD[out_ind] = w_disp * A0;
d_CD[out_ind + n_val_accum] = w_disp * A1;
d_CD[out_ind + 2 * n_val_accum] = w_disp * A2;
d_CD[out_ind + 3 * n_val_accum] = w_disp * A3;
d_CD[out_ind + 4 * n_val_accum] = w_disp * A4;
d_CD[out_ind + 5 * n_val_accum] = w_disp * A5;
d_CD[out_ind + 6 * n_val_accum] = w_disp * A6;
d_CD[out_ind + 7 * n_val_accum] = w_disp * A7;
d_CD[out_ind + 8 * n_val_accum] = w_disp * A8;
d_CD[out_ind + 9 * n_val_accum] = w_disp * A9;
d_CD[out_ind + 10 * n_val_accum] = w_disp * A10;
d_CD[out_ind + 11 * n_val_accum] = w_disp * A11;
d_CD[out_ind + 12 * n_val_accum] = w_disp * A12;
d_CD[out_ind + 13 * n_val_accum] = w_disp * A13;
d_CD[out_ind + 14 * n_val_accum] = w_disp * A14;
d_CD[out_ind + 15 * n_val_accum] = w_disp * A15;
d_CD[out_ind + 16 * n_val_accum] = w_disp * A16;
d_CD[out_ind + 17 * n_val_accum] = w_disp * A17;
d_CD[out_ind + 18 * n_val_accum] = w_disp * A18;
d_CD[out_ind + 19 * n_val_accum] = w_disp * A19;
d_CD[out_ind + 20 * n_val_accum] = w_disp * A20;
d_CD[out_ind + 21 * n_val_accum] = w_disp * A21;
d_CD[out_ind + 22 * n_val_accum] = w_disp * A22;
d_CD[out_ind + 23 * n_val_accum] = w_disp * A23;
d_CD[out_ind + 24 * n_val_accum] = w_disp * A24;
d_CD[out_ind + 25 * n_val_accum] = w_disp * A25;
d_CD[out_ind + 26 * n_val_accum] = w_disp * A26;
}