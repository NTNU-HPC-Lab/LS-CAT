#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void invalidateFlow_kernel(float *modFlowX, float *modFlowY, const float *constFlowX, const float *constFlowY, int width, int height, float cons_thres) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

if (x < width && y < height) {
int ind = __mul24(y, width) + x;
float mFX = modFlowX[ind];
float mFY = modFlowY[ind];
float cFX = constFlowX[ind];
float cFY = constFlowY[ind];

float err = (mFX - cFX) * (mFX - cFX) + (mFY - cFY) * (mFY - cFY);
err = sqrtf(err);

if (err > cons_thres) {
mFX = nanf("");
mFY = nanf("");
}

modFlowX[ind] = mFX;
modFlowY[ind] = mFY;
}
}