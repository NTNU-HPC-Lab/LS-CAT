#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deInterleave_kernel2(float *d_X_out, float *d_Y_out, char *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < width) & (y < height)) { // are we in the image?
float *data = (float *)(d_XY_in + y * pitch_in) + 2 * x;
*((float *)((char *)d_X_out + y *pitch_out) + x) = data[0];
*((float *)((char *)d_Y_out + y *pitch_out) + x) = data[1];
}
}