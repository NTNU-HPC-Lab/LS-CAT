#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blendFloatImageFloatLabelToRGBA_kernel( uchar4 *out_image, const float *in_image, const float *label, int width, int height, float lowerLim, float upperLim) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 temp;
if (x < width && y < height) {
unsigned char img =
(unsigned char)(0.5f * in_image[__mul24(y, width) + x] + 128.0f);
float val = label[__mul24(y, width) + x];

// draw everything invalid or out of lim in white
if (!isfinite(val) || (val < lowerLim) || (val > upperLim)) {
// don't blend

temp.x = img;
temp.y = img;
temp.z = img;
temp.w = 255;

} else {

// blend

temp.x = 0.6f * img;
temp.y = 0.6f * img;
temp.z = img;
temp.w = 255;
}
out_image[__mul24(y, width) + x] = temp;
}
}