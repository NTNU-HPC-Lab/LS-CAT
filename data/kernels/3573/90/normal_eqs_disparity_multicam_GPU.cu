#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_disparity_multicam_GPU( float *d_CD, float *d_disparity_compact, float4 *d_Zbuffer_normals_compact, int *d_ind_disparity_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const float *d_baseline, const int *d_n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, const int *d_pixel_ind_offset) {
int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be
// multiple of blocksize

int n_disparity = d_n_values_disparity[blockIdx.y];
int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
int start_ind = d_start_ind_disparity[blockIdx.y];

float f = d_focal_length[blockIdx.y];
float ox = d_nodal_point_x[blockIdx.y];
float oy = d_nodal_point_y[blockIdx.y];
float b = d_baseline[blockIdx.y];
int n_cols = d_n_cols[blockIdx.y];
int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

// initialize accumulators

float A0 = 0.0f, A1 = 0.0f, A2 = 0.0f, A3 = 0.0f, A4 = 0.0f, A5 = 0.0f,
A6 = 0.0f, A7 = 0.0f, A8 = 0.0f, A9 = 0.0f, A10 = 0.0f, A11 = 0.0f,
A12 = 0.0f, A13 = 0.0f, A14 = 0.0f, A15 = 0.0f, A16 = 0.0f, A17 = 0.0f,
A18 = 0.0f, A19 = 0.0f, A20 = 0.0f, A21 = 0.0f, A22 = 0.0f, A23 = 0.0f,
A24 = 0.0f, A25 = 0.0f, A26 = 0.0f;

for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

if (in_ind < n_disparity) { // is this a valid sample?

// fetch disparity, Zbuffer and normal from global memory
float disp = d_disparity_compact[in_ind + start_ind];
float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
float Zbuffer = tmp.x;
float nx = tmp.y;
float ny = tmp.z;
float nz = tmp.w;

// compute coordinates
int pixel_ind =
d_ind_disparity_Zbuffer[in_ind + start_ind] - pixel_ind_offset;

float y = floorf(__fdividef((float)pixel_ind, n_cols));
float x = (float)pixel_ind - y * n_cols;

x = __fdividef((x - ox), f);
y = -__fdividef((y - oy), f);

// reconstruct 3D point from disparity

float Zd = -(f * b) / disp;
float Xd = x * Zd;
float Yd = y * Zd;

// reconstruct 3D point from model

float Zm = Zbuffer;
float Xm = x * Zm;
float Ym = y * Zm;

/************************/
/* evaluate constraints */
/************************/

// unique values A-matrix

A0 += nx * nx;
A1 += nx * ny;
A2 += nx * nz;
A3 += Ym * nx * nz - Zm * nx * ny;
A4 += Zm * (nx * nx) - Xm * nx * nz;
A5 += -Ym * (nx * nx) + Xm * nx * ny;

A6 += ny * ny;
A7 += ny * nz;
A8 += -Zm * (ny * ny) + Ym * ny * nz;
A9 += -Xm * ny * nz + Zm * nx * ny;
A10 += Xm * (ny * ny) - Ym * nx * ny;

A11 += nz * nz;
A12 += Ym * (nz * nz) - Zm * ny * nz;
A13 += -Xm * (nz * nz) + Zm * nx * nz;
A14 += Xm * ny * nz - Ym * nx * nz;

A15 += (Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) -
Ym * Zm * ny * nz * 2.0f;
A16 += -Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny + Xm * Zm * ny * nz +
Ym * Zm * nx * nz;
A17 += -Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz + Xm * Ym * ny * nz +
Ym * Zm * nx * ny;

A18 += (Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) -
Xm * Zm * nx * nz * 2.0f;
A19 += -Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz + Xm * Ym * nx * nz +
Xm * Zm * nx * ny;

A20 += (Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) -
Xm * Ym * nx * ny * 2.0f;

// B-vector

A21 += Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny - Ym * nx * ny +
Zd * nx * nz - Zm * nx * nz;
A22 += Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny - Xm * nx * ny +
Zd * ny * nz - Zm * ny * nz;
A23 += Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz - Xm * nx * nz +
Yd * ny * nz - Ym * ny * nz;
A24 += -Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) -
Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny +
Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz;
A25 += Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) +
Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz +
Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz;
A26 += -Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) -
Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz -
Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz;
}
}

/**************************/
/* write out accumulators */
/**************************/

int out_ind =
27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

d_CD[out_ind] = A0;
d_CD[out_ind + n_val_accum] = A1;
d_CD[out_ind + 2 * n_val_accum] = A2;
d_CD[out_ind + 3 * n_val_accum] = A3;
d_CD[out_ind + 4 * n_val_accum] = A4;
d_CD[out_ind + 5 * n_val_accum] = A5;
d_CD[out_ind + 6 * n_val_accum] = A6;
d_CD[out_ind + 7 * n_val_accum] = A7;
d_CD[out_ind + 8 * n_val_accum] = A8;
d_CD[out_ind + 9 * n_val_accum] = A9;
d_CD[out_ind + 10 * n_val_accum] = A10;
d_CD[out_ind + 11 * n_val_accum] = A11;
d_CD[out_ind + 12 * n_val_accum] = A12;
d_CD[out_ind + 13 * n_val_accum] = A13;
d_CD[out_ind + 14 * n_val_accum] = A14;
d_CD[out_ind + 15 * n_val_accum] = A15;
d_CD[out_ind + 16 * n_val_accum] = A16;
d_CD[out_ind + 17 * n_val_accum] = A17;
d_CD[out_ind + 18 * n_val_accum] = A18;
d_CD[out_ind + 19 * n_val_accum] = A19;
d_CD[out_ind + 20 * n_val_accum] = A20;
d_CD[out_ind + 21 * n_val_accum] = A21;
d_CD[out_ind + 22 * n_val_accum] = A22;
d_CD[out_ind + 23 * n_val_accum] = A23;
d_CD[out_ind + 24 * n_val_accum] = A24;
d_CD[out_ind + 25 * n_val_accum] = A25;
d_CD[out_ind + 26 * n_val_accum] = A26;
}