#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertFloatToRGBAbinary_kernel(uchar4 *out_image, const float *in_image, int width, int height, float lowerLim, float upperLim) {
const int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
const int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
uchar4 temp;
if (x < width && y < height) {
float val = in_image[__mul24(y, width) + x];

// draw everything invalid or out of lim in white
if (!isfinite(val) || (val < lowerLim) || (val > upperLim)) {
temp.x = 255;
temp.y = 255;
temp.z = 255;
temp.w = 255;
} else {
temp.x = 0.0f;
temp.y = 0.0f;
temp.z = 0.0f;
temp.w = 0.0f;
}
out_image[__mul24(y, width) + x] = temp;
}
}