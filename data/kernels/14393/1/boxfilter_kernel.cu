#include "hip/hip_runtime.h"
#include "includes.h"



// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void boxfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest, int bw, int bh)
{
// Calculate our pixel's location
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

// Variables to store the sum
int count = 0;
float sum = 0.0;

// Do the blur operation by summing the surround pixels
for(int j = -(bh/2); j <= (bh/2); j++)
{
for(int i = -(bw/2); i <= (bw/2); i++)
{
// Verify that this offset is within the image boundaries
if((x+i) < iw && (x+i) >= 0 && (y+j) < ih && (y+j) >= 0)
{
sum += (float) source[((y+j) * iw) + (x+i)];
count++;
}
}
}

// Average the sum
sum /= (float) count;
dest[(y * iw) + x] = (unsigned char) sum;
}