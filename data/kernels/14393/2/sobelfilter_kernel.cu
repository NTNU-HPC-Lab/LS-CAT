#include "hip/hip_runtime.h"
#include "includes.h"



// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void sobelfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest)
{
// Calculate our pixel's location
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

// Operate only if we are in the correct boundaries
if(x > 0 && x < iw - 1 && y > 0 && y < ih - 1)
{
int gx = -source[iw*(y-1)+(x-1)] + source[iw*(y-1)+(x+1)] +
-2*source[iw*(y)+(x-1)] + 2*source[iw*(y)+(x+1)] +
-source[iw*(y+1)+(x-1)] + source[iw*(y+1)+(x+1)];
int gy = -source[iw*(y-1)+(x-1)] - 2*source[iw*(y-1)+(x)]
-source[iw*(y-1)+(x+1)] +
source[iw*(y+1)+(x-1)] + 2*source[iw*(y+1)+(x)] +
source[iw*(y+1)+(x+1)];
dest[iw*y+x] = (int) sqrt((float)(gx)*(float)(gx) + (float)(gy)*(float)(gy));
}
}