#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, ssize_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
// Calculate our pixel's location
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

float sum = 0.0f;
int   pWidth = kWidth/2;
int   pHeight = kHeight/2;

// Execute for valid pixels
if(x >= pWidth+paddingX &&
y >= pHeight+paddingY &&
x < (gridDim.x * blockDim.x)-pWidth-paddingX &&
y < (gridDim.y *blockDim.y )-pHeight-paddingY)
{
for(int j = -pHeight; j <= pHeight; j++)
{
for(int i = -pWidth; i <= pWidth; i++)
{
// Sample the weight for this location
int ki = (i+pWidth);
int kj = (j+pHeight);
float w  = convolutionKernel[(kj * kWidth) + ki + kOffset];


sum += w * float(source[((y+j) * width) + (x+i)]);
}
}
}

// Average the sum
destination[(y * width) + x] = (unsigned char) sum;
}