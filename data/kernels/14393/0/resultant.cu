#include "hip/hip_runtime.h"
#include "includes.h"



// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void resultant(unsigned char *a, unsigned char *b, unsigned char *c)
{
int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

float opposite_side = float(a[idx]);
float adjacent_side = float(b[idx]);

// Figure out the hypotenuse
c[idx] = (unsigned char) sqrtf((opposite_side + adjacent_side)*(opposite_side + adjacent_side ) - (2 * opposite_side * adjacent_side));
if ( c[idx] > 15 && c [idx -1] != 148 )
c[idx] = 148;
else
c[idx] = 0;

}