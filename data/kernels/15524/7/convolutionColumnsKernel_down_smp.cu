#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionColumnsKernel_down_smp( float *d_Dst, float *d_Src, int imageW, int imageH, int n_imageH, int pitch, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the upper halo edge
const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * 2 * COLUMNS_RESULT_STEPS - Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
const int baseY1 = (blockIdx.y * COLUMNS_RESULT_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;

if (baseX < imageW)
{
d_Src += baseY * pitch + baseX;
d_Dst += baseY1 * pitch + baseX;

//Upper halo
#pragma unroll
for (int i = 0; i < Halo_steps; i++)
{
s_Data[(threadIdx.x*(2 * COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y >= 0) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Lower halo + Main data
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + 2 * COLUMNS_RESULT_STEPS + Halo_steps; i++)
{
s_Data[(threadIdx.x*(2 * COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < imageH) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = 0; i < COLUMNS_RESULT_STEPS; ++i)
{
float sum = 0;

if (baseY1 + i * COLUMNS_BLOCKDIM_Y < n_imageH)
{
#pragma unroll
for (int j = -filter_Rad; j <= filter_Rad; ++j)
{
sum += c_Kernel[filter_Rad - j] * s_Data[(threadIdx.x*(2 * COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + 2 * threadIdx.y + 2 * i * COLUMNS_BLOCKDIM_Y + Halo_steps * COLUMNS_BLOCKDIM_Y + j];
}

d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
}
}
}
}