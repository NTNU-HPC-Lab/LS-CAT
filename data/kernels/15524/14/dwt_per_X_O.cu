#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dwt_per_X_O(float *d_ip, int rows, int cols, int cA_cols, int filt_len, int Halo_steps, float *d_cL, float *d_cH)
{
extern __shared__ float s_Data[];

//Offset to the left halo edge
const int baseX = (blockIdx.x * 2 * X_RESULT_STEPS - Halo_steps) * X_BLOCKDIM_X + threadIdx.x;
const int baseX1 = (blockIdx.x * X_RESULT_STEPS) * X_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * X_BLOCKDIM_Y + threadIdx.y;

if (baseY < rows) {

d_ip += baseY * cols + baseX;
d_cL += baseY * cA_cols + baseX1;
d_cH += baseY * cA_cols + baseX1;

//Loading data to shared memory

//Load Left Halo
#pragma unroll
for (int i = 0; i < Halo_steps; i++)
{
if (baseX + i * X_BLOCKDIM_X == -1) s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = d_ip[cols - 1];

else s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = (baseX + i * X_BLOCKDIM_X >= 0) ? d_ip[i * X_BLOCKDIM_X] : d_ip[i * X_BLOCKDIM_X + cols + 1];
}

// main data and Load right halo
#pragma unroll

for (int i = Halo_steps; i < Halo_steps + 2 * X_RESULT_STEPS + Halo_steps; i++)
{
if (baseX + i * X_BLOCKDIM_X == cols) s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = d_ip[i * X_BLOCKDIM_X - 1];

else s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = ((baseX + i * X_BLOCKDIM_X) < cols) ? d_ip[i * X_BLOCKDIM_X] : d_ip[i * X_BLOCKDIM_X - cols - 1];
}

//Compute and store results
__syncthreads();


#pragma unroll

for (int i = 0; i < X_RESULT_STEPS; i++)
{
if ((baseX1 + i * X_BLOCKDIM_X < cA_cols))
{
float sum_cL = 0, sum_cH = 0;

int l2 = filt_len / 2;

for (int l = 0; l < filt_len; ++l)
{
sum_cL += c_lpd[l] * s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x * 2 + Halo_steps*X_BLOCKDIM_X + 2 * i * X_BLOCKDIM_X + l2 - l]; //l2-l is to select the right center pixels with odd and even sized filters
sum_cH += c_hpd[l] * s_Data[(threadIdx.y*(2 * X_RESULT_STEPS + 2 * Halo_steps)*X_BLOCKDIM_X) + threadIdx.x * 2 + Halo_steps*X_BLOCKDIM_X + 2 * i * X_BLOCKDIM_X + l2 - l];
}
d_cL[i * X_BLOCKDIM_X] = sum_cL;
d_cH[i * X_BLOCKDIM_X] = sum_cH;
}
}
}
}