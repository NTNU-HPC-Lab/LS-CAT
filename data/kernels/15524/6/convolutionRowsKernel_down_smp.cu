#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionRowsKernel_down_smp( float *d_Dst, float *d_Src, int imageW, int n_imageW, int imageH, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the left halo edge
const int baseX = (blockIdx.x * 2 * ROWS_RESULT_STEPS - Halo_steps) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseX1 = (blockIdx.x * ROWS_RESULT_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

if (baseY < imageH)
{
d_Src += baseY * imageW + baseX;
d_Dst += baseY * n_imageW + baseX1;

//Load left halo
#pragma unroll
for (int i = 0; i < Halo_steps; ++i)
{
s_Data[(threadIdx.y*(2 * ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X >= 0) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Load right halo and main data
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + 2 * ROWS_RESULT_STEPS + Halo_steps; ++i)
{
s_Data[(threadIdx.y*(2 * ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X < imageW) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();

#pragma unroll
for (int i = 0; i < ROWS_RESULT_STEPS; ++i)
{
float sum = 0;
if (baseX1 + i * ROWS_BLOCKDIM_X < n_imageW)
{
#pragma unroll
for (int j = -filter_Rad; j <= filter_Rad; ++j)
{
sum += c_Kernel[filter_Rad - j] * s_Data[(threadIdx.y*(2 * ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + (Halo_steps + 2 * i) * ROWS_BLOCKDIM_X + threadIdx.x * 2 + j];
}

d_Dst[i * ROWS_BLOCKDIM_X] = sum;

}
}
}
}