#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dwt_per_Y_E(float *d_ip, int rows, int cols, int cA_rows, int filt_len, int Halo_steps, float *d_cL, float *d_cH) {
extern __shared__ float s_Data[];

//Offset to the upper halo edge
const int baseX = blockIdx.x * Y_BLOCKDIM_X + threadIdx.x;
const int baseY = ((blockIdx.y * 2 * Y_RESULT_STEPS) - Halo_steps) * Y_BLOCKDIM_Y + threadIdx.y;
const int baseY1 = (blockIdx.y * Y_RESULT_STEPS) * Y_BLOCKDIM_Y + threadIdx.y;

if (baseX < cols)
{
d_ip += baseY * cols + baseX;
d_cL += baseY1 * cols + baseX;
d_cH += baseY1 * cols + baseX;

//Loading data to shared memory
//Upper halo
#pragma unroll

for (int i = 0; i < Halo_steps; i++)
{
s_Data[(threadIdx.x*(2 * Y_RESULT_STEPS + 2 * Halo_steps) *Y_BLOCKDIM_Y) + threadIdx.y + i * Y_BLOCKDIM_Y] = (baseY + i * Y_BLOCKDIM_Y >= 0) ? d_ip[i * Y_BLOCKDIM_Y * cols] : d_ip[(i * Y_BLOCKDIM_Y * cols) + (rows*cols)];
}

//Lower halo + Main data
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + 2 * Y_RESULT_STEPS + Halo_steps; i++)
{
s_Data[(threadIdx.x*(2 * Y_RESULT_STEPS + 2 * Halo_steps) *Y_BLOCKDIM_Y) + threadIdx.y + i * Y_BLOCKDIM_Y] = (baseY + i * Y_BLOCKDIM_Y < rows) ? d_ip[i * Y_BLOCKDIM_Y * cols] : d_ip[(i * Y_BLOCKDIM_Y * cols) - (rows*cols)];
}

__syncthreads();

//Compute and store results
#pragma unroll
for (int i = 0; i < Y_RESULT_STEPS; i++)
{
if ((baseY1 + i * Y_BLOCKDIM_Y < cA_rows)) {
int l2 = filt_len / 2;

float sum_cL = 0, sum_cH = 0;

for (int l = 0; l < filt_len; ++l)
{
sum_cL += c_lpd[l] * s_Data[(threadIdx.x*(2 * Y_RESULT_STEPS + 2 * Halo_steps) *Y_BLOCKDIM_Y) + 2 * threadIdx.y + 2 * i * Y_BLOCKDIM_Y + Halo_steps * Y_BLOCKDIM_Y + l2 - l];
sum_cH += c_hpd[l] * s_Data[(threadIdx.x*(2 * Y_RESULT_STEPS + 2 * Halo_steps) *Y_BLOCKDIM_Y) + 2 * threadIdx.y + 2 * i * Y_BLOCKDIM_Y + Halo_steps * Y_BLOCKDIM_Y + l2 - l];
}

d_cL[i * Y_BLOCKDIM_Y * cols] = sum_cL;
d_cH[i * Y_BLOCKDIM_Y * cols] = sum_cH;
}
}
}

}