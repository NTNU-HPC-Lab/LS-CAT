#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void add(float *d_dst, float*d_src_1, int len) {


int baseX = blockIdx.x * blockDim.x + threadIdx.x;

if (baseX < len)
{
d_dst[baseX] = d_dst[baseX] + d_src_1[baseX];
}

}