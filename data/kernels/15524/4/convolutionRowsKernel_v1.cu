#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionRowsKernel_v1( float *d_Dst, float *d_Src, int imageW, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the left halo edge
const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - Halo_steps) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

d_Src += baseY * imageW + baseX;
d_Dst += baseY * imageW + baseX;

//Load main data
/*#pragma unroll

for (int i = Halo_steps; i < Halo_steps + ROWS_RESULT_STEPS; i++)
{
s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
}*/

//Load left halo
#pragma unroll
for (int i = 0; i < Halo_steps; i++)
{
s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X >= 0) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Load right halo and main data
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + ROWS_RESULT_STEPS + Halo_steps; i++)
{
s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X < imageW) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + ROWS_RESULT_STEPS; i++)
{
float sum = 0;
if (baseX + i * ROWS_BLOCKDIM_X < imageW)
{
#pragma unroll
for (int j = -filter_Rad; j <= filter_Rad; j++)
{
sum += c_Kernel[filter_Rad - j] * s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X + j];
}

d_Dst[i * ROWS_BLOCKDIM_X] = sum;

}
}
}