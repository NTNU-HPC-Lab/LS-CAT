#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void idwt_per_Y_1(float *d_dst, float *src_A, float *src_D, int rows, int cols, int next_rows, int filt_len, int halo) {

extern __shared__ float s_Data[];

//Offset to the upper halo edge
const int baseX = blockIdx.x * I_Y_BLOCKDIM_X + threadIdx.x;
const int baseY = ((blockIdx.y * I_Y_RESULT_STEPS) - halo) * I_Y_BLOCKDIM_Y + threadIdx.y;

int l2 = filt_len / 2;

if (baseX < cols)
{
src_A += baseY * cols + baseX;
src_D += baseY * cols + baseX;
d_dst += (2 * baseY - l2 + 1) * cols + baseX;

//Loading data to shared memory
//Upper halo
#pragma unroll

for (int i = 0; i < halo; i++)
{
s_Data[(threadIdx.x*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (baseY + i * I_Y_BLOCKDIM_Y >= 0) ? src_A[i * I_Y_BLOCKDIM_Y * cols] : src_A[(i * I_Y_BLOCKDIM_Y * cols) + (rows*cols)];
s_Data[((threadIdx.x + I_Y_BLOCKDIM_X)*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (baseY + i * I_Y_BLOCKDIM_Y >= 0) ? src_D[i * I_Y_BLOCKDIM_Y * cols] : src_D[(i * I_Y_BLOCKDIM_Y * cols) + (rows*cols)];
}

//Lower halo + Main data
#pragma unroll
for (int i = halo; i < halo + I_Y_RESULT_STEPS + halo; i++)
{
s_Data[(threadIdx.x*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (baseY + i * I_Y_BLOCKDIM_Y < rows) ? src_A[i * I_Y_BLOCKDIM_Y * cols] : src_A[(i * I_Y_BLOCKDIM_Y * cols) - (rows*cols)];
s_Data[((threadIdx.x + I_Y_BLOCKDIM_X)*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (baseY + i * I_Y_BLOCKDIM_Y < rows) ? src_D[i * I_Y_BLOCKDIM_Y * cols] : src_D[(i * I_Y_BLOCKDIM_Y * cols) - (rows*cols)];
}

__syncthreads();
#pragma unroll
for (int i = halo; i < I_Y_RESULT_STEPS + halo; i++)
{
int pos_y = 2 * baseY + 2 * i * I_Y_BLOCKDIM_Y;

if (pos_y + 1 < (2 * rows + filt_len - 2)) {

float temp_1 = 0, temp_2 = 0;


for (int l = 0; l < l2; ++l)
{
int t = 2 * l;

temp_1 += c_lpr[t] * s_Data[(threadIdx.x*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i* I_Y_BLOCKDIM_Y - l]
+ c_hpr[t] * s_Data[((threadIdx.x + I_Y_BLOCKDIM_X)*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y - l];
temp_2 += c_lpr[t + 1] * s_Data[(threadIdx.x*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y - l]
+ c_hpr[t + 1] * s_Data[((threadIdx.x + I_Y_BLOCKDIM_X)*(I_Y_RESULT_STEPS + 2 * halo) *I_Y_BLOCKDIM_Y) + threadIdx.y + i * I_Y_BLOCKDIM_Y - l];
}

if ((pos_y >= l2 - 1) && (pos_y < next_rows + l2 - 1)) d_dst[2 * i * I_Y_BLOCKDIM_Y * cols] = temp_1;
if ((pos_y + 1 >= l2 - 1) && (pos_y + 1 < next_rows + l2 - 1)) d_dst[(2 * i * I_Y_BLOCKDIM_Y + 1) * cols] = temp_2;
}
}

}
}