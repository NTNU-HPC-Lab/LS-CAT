#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionColumnsKernel_up_smp( float *d_Dst, float *d_Src, int imageW, int imageH, int n_imageH, int pitch, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the upper halo edge
const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;

if (baseX < imageW)
{
d_Src += baseY * pitch + baseX;
d_Dst += 2 * baseY * pitch + baseX;

//Upper halo
//#pragma unroll
for (int i = 0; i < Halo_steps; i++)
{
s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y >= 0) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Lower halo + Main data
//#pragma unroll
for (int i = Halo_steps; i < Halo_steps + COLUMNS_RESULT_STEPS + Halo_steps; i++)
{
s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < imageH) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Compute and store results
__syncthreads();
//#pragma unroll
for (int i = Halo_steps; i < COLUMNS_RESULT_STEPS + Halo_steps; ++i)
{
int Pos_y = 2 * baseY + (2 * i) * COLUMNS_BLOCKDIM_Y;

if (Pos_y < n_imageH)
{
float sum_1 = 0.0f, sum_2 = 0.0f;

//#pragma unroll
for (int l = -(filter_Rad / 2); l <= filter_Rad / 2; ++l)
{
int t = 2 * l;

float temp = s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y + l];

sum_1 += c_Kernel[filter_Rad + t] * temp * 2.0f;
sum_2 += c_Kernel[filter_Rad + t - 1] * temp * 2.0f;
}

sum_2 += c_Kernel[2 * filter_Rad] * 2.0f * s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y + filter_Rad / 2 + 1];

d_Dst[2 * i * COLUMNS_BLOCKDIM_Y * pitch] = sum_1;
if (Pos_y + 1 < n_imageH)d_Dst[2 * i * COLUMNS_BLOCKDIM_Y * pitch + pitch] = sum_2;

}

}
}
}