#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dwt_compare(float *d_ip_v, float *d_ip_ir, int len) {

const int X = blockIdx.x * blockDim.x + threadIdx.x;

if (X < len)
{
d_ip_v[X] = (abs(d_ip_v[X]) > abs(d_ip_ir[X])) ? d_ip_v[X] : d_ip_ir[X];
}

}