#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void subtract(float *d_dst, float*d_src_1, float* d_src_2, int len) {


int baseX = blockIdx.x * blockDim.x + threadIdx.x;

if (baseX < len)
{
d_dst[baseX] = d_src_1[baseX] - d_src_2[baseX];
}

}