#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionRowsKernel_up_smp( float *d_Dst, float *d_Src, int imageW, int n_imageW, int imageH, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the left halo edge
const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - Halo_steps) * ROWS_BLOCKDIM_X + threadIdx.x;
const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
const int baseX1 = blockIdx.x * ROWS_RESULT_STEPS * 2 * ROWS_BLOCKDIM_X + 2 * threadIdx.x;

if (baseY < imageH)
{
d_Src += baseY * imageW + baseX;
d_Dst += baseY * n_imageW + baseX1;

//Load left halo
//#pragma unroll
for (int i = 0; i < Halo_steps; ++i)
{
s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X >= 0) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Load right halo and main data
//#pragma unroll
for (int i = Halo_steps; i < Halo_steps + ROWS_RESULT_STEPS + Halo_steps; ++i)
{
s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX + i * ROWS_BLOCKDIM_X < imageW) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
}

//Compute and store results
__syncthreads();

//#pragma unroll
for (int i = Halo_steps; i < Halo_steps + ROWS_RESULT_STEPS; ++i)
{
int pos_x = (baseX1 + 2 * (i - Halo_steps) * ROWS_BLOCKDIM_X);

if (pos_x < n_imageW)
{
float sum_1 = 0.0f, sum_2 = 0.0f;

//#pragma unroll
for (int l = -(filter_Rad / 2); l <= filter_Rad / 2; ++l)
{
int t = 2 * l;

float temp = s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X + l];
sum_1 += c_Kernel[filter_Rad + t] * temp *2.0f;
sum_2 += c_Kernel[filter_Rad + t - 1] * temp *2.0f;

}

sum_2 += c_Kernel[2 * filter_Rad] * 2.0f * s_Data[(threadIdx.y*(ROWS_RESULT_STEPS + 2 * Halo_steps)*ROWS_BLOCKDIM_X) + threadIdx.x + i * ROWS_BLOCKDIM_X + filter_Rad / 2 + 1];

d_Dst[2 * (i - Halo_steps)* ROWS_BLOCKDIM_X] = sum_1;
if (pos_x + 1 < n_imageW) d_Dst[2 * (i - Halo_steps) * ROWS_BLOCKDIM_X + 1] = sum_2;
}
}
}
}