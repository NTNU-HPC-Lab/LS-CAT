#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void average(float *d_ip_v, float *d_ip_ir, int app_len) {

const int X = blockIdx.x * blockDim.x + threadIdx.x;

if (X < app_len)
{
d_ip_v[X] = (d_ip_v[X] + d_ip_ir[X]) / 2;
}

}