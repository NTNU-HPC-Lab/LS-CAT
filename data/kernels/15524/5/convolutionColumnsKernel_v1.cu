#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionColumnsKernel_v1( float *d_Dst, float *d_Src, int imageW, int imageH, int pitch, int filter_Rad, int Halo_steps )
{
extern __shared__ float s_Data[];

//Offset to the upper halo edge
const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
d_Src += baseY * pitch + baseX;
d_Dst += baseY * pitch + baseX;

/*	//Main data
#pragma unroll

for (int i = Halo_steps; i < Halo_steps + COLUMNS_RESULT_STEPS; i++)
{
s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS+2*Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
}*/

//Upper halo
#pragma unroll
for (int i = 0; i < Halo_steps; i++)
{
s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Lower halo + Main data
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + COLUMNS_RESULT_STEPS + Halo_steps; i++)
{
s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < imageH) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
}

//Compute and store results
__syncthreads();
#pragma unroll
for (int i = Halo_steps; i < Halo_steps + COLUMNS_RESULT_STEPS; i++)
{
float sum = 0;

if (baseY + i * COLUMNS_BLOCKDIM_Y < imageH)
{
#pragma unroll
for (int j = -filter_Rad; j <= filter_Rad; j++)
{
sum += c_Kernel[filter_Rad - j] * s_Data[(threadIdx.x*(COLUMNS_RESULT_STEPS + 2 * Halo_steps) *COLUMNS_BLOCKDIM_Y) + threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
}

d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
}
}
}