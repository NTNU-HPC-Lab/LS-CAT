#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void d_transpose(float *odata, float *idata, int width, int height)
{
__shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

// read the matrix tile into shared memory
unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

if ((xIndex < width) && (yIndex < height))
{
unsigned int index_in = yIndex * width + xIndex;
block[threadIdx.y][threadIdx.x] = idata[index_in];
}

__syncthreads();

// write the transposed matrix tile to global memory
xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

if ((xIndex < height) && (yIndex < width))
{
unsigned int index_out = yIndex * height + xIndex;
odata[index_out] = block[threadIdx.x][threadIdx.y];
}
}