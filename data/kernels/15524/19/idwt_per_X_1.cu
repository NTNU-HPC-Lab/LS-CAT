#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void idwt_per_X_1(float *d_dst, float *src_A, float *src_D, int rows, int cols, int next_cols, int filt_len, int halo) {

extern __shared__ float s_Data[];

//Offset to the left halo edge
const int baseX = ((blockIdx.x * I_X_RESULT_STEPS) - halo) * I_X_BLOCKDIM_X + threadIdx.x; // even if the last pixel of cols+l2-1 takes a new block, rows can be maintained
const int baseY = blockIdx.y * I_X_BLOCKDIM_Y + threadIdx.y;

const int baseX1 = blockIdx.x * I_X_RESULT_STEPS * 2 * I_X_BLOCKDIM_X + 2 * threadIdx.x;

if (baseY < rows) {

src_A += baseY * cols + baseX;
src_D += baseY * cols + baseX;
d_dst += baseY * next_cols + baseX1;//To compensate the halo

int l2 = filt_len / 2;

//Loading data to shared memory
#pragma unroll
for (int i = 0; i < halo; i++)
{
s_Data[(threadIdx.y*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X] = (baseX + i * I_X_BLOCKDIM_X >= 0) ? src_A[i * I_X_BLOCKDIM_X] : src_A[i * I_X_BLOCKDIM_X + cols];
s_Data[((threadIdx.y + I_X_BLOCKDIM_Y)*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X] = (baseX + i * I_X_BLOCKDIM_X >= 0) ? src_D[i * I_X_BLOCKDIM_X] : src_D[i * I_X_BLOCKDIM_X + cols];
}

// main data and Load right halo
#pragma unroll

for (int i = halo; i < halo + I_X_RESULT_STEPS + halo; i++)
{

s_Data[(threadIdx.y*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X] = ((baseX + i * I_X_BLOCKDIM_X) < cols) ? src_A[i * I_X_BLOCKDIM_X] : src_A[i * I_X_BLOCKDIM_X - cols];
s_Data[((threadIdx.y + I_X_BLOCKDIM_Y)*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X] = ((baseX + i * I_X_BLOCKDIM_X) < cols) ? src_D[i * I_X_BLOCKDIM_X] : src_D[i * I_X_BLOCKDIM_X - cols];

}

//Compute and store results
__syncthreads();

#pragma unroll
for (int i = halo; i < halo + I_X_RESULT_STEPS; i++)
{
int pos_x = (baseX1 + 2 * (i - halo) * I_X_BLOCKDIM_X);

if ((pos_x + 1) < (2 * cols + filt_len - 2)) {

float temp_1 = 0, temp_2 = 0;

for (int l = 0; l < l2; ++l)
{
int t = 2 * l;

temp_1 += c_lpr[t] * s_Data[(threadIdx.y*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l]
+ c_hpr[t] * s_Data[((threadIdx.y + I_X_BLOCKDIM_Y)*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l];
temp_2 += c_lpr[t + 1] * s_Data[(threadIdx.y*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l]
+ c_hpr[t + 1] * s_Data[((threadIdx.y + I_X_BLOCKDIM_Y)*(I_X_RESULT_STEPS + 2 * halo)*I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l];
}

if ((pos_x >= l2 - 1) && (pos_x < (next_cols + l2 - 1))) d_dst[2 * (i - halo) * I_X_BLOCKDIM_X - l2 + 1] = temp_1;
if ((pos_x + 1 >= l2 - 1) && (pos_x + 1 < (next_cols + l2 - 1))) d_dst[2 * (i - halo) * I_X_BLOCKDIM_X - l2 + 2] = temp_2;
}

}
}
}