#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void compare(float *d_ip_v, float *d_ip_ir, int len) {

const int X = blockIdx.x * blockDim.x + threadIdx.x;

if (X < len)
{
d_ip_v[X] = (abs(d_ip_v[X]) > abs(d_ip_ir[X])) ? d_ip_v[X] : d_ip_ir[X];
}

}