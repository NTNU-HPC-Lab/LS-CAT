#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dwt_average(float *d_ip_v, float *d_ip_ir, int app_len) {

const int X = blockIdx.x * blockDim.x + threadIdx.x;

if (X < app_len)
{
d_ip_v[X] = (d_ip_v[X] + d_ip_ir[X]) / 2;
}

}