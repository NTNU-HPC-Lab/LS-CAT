#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReferenceGemm_kernel( int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc) {

int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

if (i < M && j < N) {
float accumulator = 0;

for (int k = 0; k < K; ++k) {
accumulator += A[i + k * lda] * B[k + j * ldb];
}

C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
}
}