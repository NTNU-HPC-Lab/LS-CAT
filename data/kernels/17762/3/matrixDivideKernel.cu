#include "hip/hip_runtime.h"
#include "includes.h"
//Author: Adriel Kim
//6-27-2020
//Updated 7-5-2020
/*
Desc: Basic 2D matrix operations - element-wise addition, subtraction, multiplication, and division.

To do:
- Use vector instead of array?
- Be able to test for varying sizes of images. (For now we manually define with constant N)
- Add timer to compare CPU and GPU implementations
- Double check if all memory is freed
- Optimize by eliminating redundant calculations
- Test code on department servers
*/


//define imin(a,b)  (a<b?a:b)//example of ternary operator in c++
//4176,2048
#define R 4176
#define C 2048
#define N (R*C)//# of elements in matrices
const int threadsPerBlock = 1024;//threads in a block. A chunk that shares the same shared memory.
const int blocksPerGrid = 8352;//imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);//this will be our output array size for sumKernel.

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

//any advantages with mapping directly to strucutre of matrix? We're just representing 2D matrix using 1D array...
//it would be difficult to do the above since we want the operations to occur over abitrarily large matrices
//this can definitely be optimzied by elminating redundant calculations

//---------------------------------------------------------------------------------
__global__ void matrixDivideKernel(double* c, const double* a, const double* b) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < N) {
c[tid] = (a[tid]/b[tid]);
tid += blockDim.x * gridDim.x;
}
}