#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void cruza(unsigned int n, unsigned int np, int *cidadesAle, int *pop, int *newPop, int *poolPais, int *mutacoes) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

int paiA, paiB, copiaPai, crossover, mutar, pontoMutar;

for (int i=index; i<n; i+=stride) {
copiaPai = cidadesAle[i*4];
crossover = cidadesAle[(i+1)*4] % np;
mutar = cidadesAle[(i+2)*4];
pontoMutar = cidadesAle[(i+3)*4] % np;
paiA = poolPais[i];
paiB = poolPais[i+1];

if (copiaPai < ELITE) {
for (int j=0; j<np; j++) {
newPop[(i*np) + j] = pop[(paiA*np) + j];
continue;
}
}
for(int j=0;j<np;j++)
{
newPop[(i*np) + j] = pop[(paiA*np) + j];
}
int t=0, aux=0, crossoverSup;
crossoverSup=(crossover +mutacoes[i]>MAX)?(MAX):(crossover +mutacoes[i]);
for(int j=crossover; j<crossoverSup;j++)
{
t=0;
while(newPop[(i*np) +t]!=pop[(paiB*np) + j])
{
t++;
}
aux = newPop[i*np+j];
newPop[i*np+j] = newPop[i*np+t];
newPop[i*np+t] = aux;

}

if (mutar < MUT) {
int mut = (mutacoes[i]>MAX)?(MAX):((mutacoes[i]<MIN)?(MIN):(mutacoes[i]));
t=0;
while(newPop[(i*np) +t]!=mut)
{
t++;
}
aux = newPop[i*np+pontoMutar];
newPop[i*np+pontoMutar] = newPop[i*np+t];
newPop[i*np+t] = aux;

}

}

}