#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void escolhePais(unsigned int n, unsigned int np, int *paisAle, double *fitness, int *pool) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;


for (int i=index; i<n; i+=stride) {
double best = 10000.0;
int best_index = -1;
int idx;

for (int j=0; j<QUANT_PAIS_AVALIA; j++) {
idx = paisAle[i*QUANT_PAIS_AVALIA+j];
if (fitness[idx] < best) {
best = fitness[idx];
best_index = idx;
}
}
pool[i] = best_index;
}
}