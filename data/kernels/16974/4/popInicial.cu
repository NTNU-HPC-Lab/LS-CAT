#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void popInicial(unsigned int n,unsigned int np,int* v, int* genes, int* ale)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i=index; i<n; i+=stride)
{
for(int j=0; j<np; j++)
{
int p = (ale[i*np+j]<j)?j:ale[i*np+j];
v[i*np+j] = genes[i*np+p];
int aux = genes[i*np+j];
genes[i*np +j] = genes[i*np+p];
genes[i*np+p]=aux;
}
}
}