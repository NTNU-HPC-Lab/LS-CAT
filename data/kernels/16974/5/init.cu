#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void init(unsigned int seed, hiprandState_t* states) {
int idx = threadIdx.x+blockDim.x*blockIdx.x;

hiprand_init(seed, idx, 0,  &states[idx]);
}