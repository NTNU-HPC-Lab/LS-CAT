#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void preencheGenes(unsigned int n,unsigned int np, int* genes)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i=index; i<n; i+=stride)
for(int j=0;j<np;j++)
genes[i*np+j]=j;

}