#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust


//Biblioteca cuRAND


//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void setRandom(hiprandState_t* states, int* numbers, int max) {
int idx = threadIdx.x+blockDim.x*blockIdx.x;

numbers[idx] = hiprand(&states[idx]) % max;

}