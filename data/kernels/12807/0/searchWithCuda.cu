#include "hip/hip_runtime.h"
#include "includes.h"

int answersNumber;
int categoriesNumber;
int atribsNumber;

/**
* Funkcja wykonywana na karcie graficznej - kazdy watek sprawdza czy jego atrybut z atribsValues to ten sam co w query. Jesli tak, przepisuje do
* tablicy wynikowej prawdopodobiestwa dla kazdej jego odpowiedzi
* @param query - zapytanie uzytkownika w postacie zlepionych stringow
* @param atribsValues - tablica wszystkich atrybutow
* @param possibilities - tablica wszystkich prawdopodobienstw
* @param queryPrefix - tablica sum prefiksowych dlugosci slow w query
* @param atribsPrefix - j.k. dla atribsValues
* @param answersNumber - liczba mozliwych odpowiedzi
* @param categoriesNumber - liczba kategorii
* @param atribsNumber - liczba wszystkich atrybutow
* @param resultPossibilities - tablica prawdopodobienstw atrybutow z zapytania dla wszystkich mozliwych odpowiedzi
*/

__global__ void searchWithCuda(double *resultPossibilities, char *query, char *atribsValues, double *possibilities, int *queryPrefix, int *atribsPrefix, int *answersNumber, int *categoriesNumber, int *atribsNumber)
{
int category_id = blockIdx.x;	// categories
int atrib_id = blockIdx.y;	// atribs

// znajdz poczatek lancucha znakow atrybutu w zapytaniu i w atribsValue
char *queryAtrib = query + queryPrefix[category_id];
int queryAtribLength = queryPrefix[category_id + 1] - queryPrefix[category_id];

char *currAtrib = atribsValues + atribsPrefix[atrib_id];
int currAtribLength = atribsPrefix[atrib_id + 1] - atribsPrefix[atrib_id];

if (queryAtribLength == currAtribLength)
{
bool equal = true;
for (int i = 0; i < queryAtribLength; ++i)
{
if (queryAtrib[i] != currAtrib[i])
{
equal = false;
break;
}
}
if (equal)	// przypisz odpowiednie prawdopodobienstwa
{
for (int i = 0; i < *answersNumber; ++i)
{
resultPossibilities[*categoriesNumber*i + category_id] = possibilities[*atribsNumber*i + atrib_id];	// na razie tylko dla jednej odpowiedzi
}
}
}
}