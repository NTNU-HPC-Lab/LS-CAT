#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel( float* g_idata, float* g_odata)
{
float result=1;
// read two values
float val1 = g_idata[0];
float val2 = g_idata[1];

// place loop/unrolled loop here to do a bunch of multiply add ops
// make sure you use results, so compiler does not optomize out
result = val2 + (result * val1);

g_odata[0] = result;
}