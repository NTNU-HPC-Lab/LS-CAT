#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_kernel(float *g_odata, float *g_idata, int n)
{
// the size is determined by the host application
extern  __shared__  float sdata[];

// access thread id
const unsigned int tid = threadIdx.x;
// access number of threads in this block
//const unsigned int num_threads = blockDim.x;

// read in input data from global memory
sdata[2*tid] = g_idata[2*tid];
sdata[2*tid+1] = g_idata[2*tid+1];

//  printf ("KERNEL: sdata[%d] = %f\n", (2*tid), sdata[2*tid]);
//  printf ("KERNEL: sdata[%d] = %f\n", (2*tid), sdata[2*tid+1]);
__syncthreads();

// perform some computations
sdata[2*tid] = sdata[2*tid] + sdata[2*tid+1];
__syncthreads();

g_odata[tid]   = sdata[tid];

}