#include "hip/hip_runtime.h"
#include "includes.h"
// Header files

// Routine to check error
__global__ void sum(int *A , int *B, int *C, long long N)
{
long long idx = blockIdx.x * blockDim.x + threadIdx.x ;
if(idx  < N)
{
C[idx] = A[idx] + B[idx] ;
}
}