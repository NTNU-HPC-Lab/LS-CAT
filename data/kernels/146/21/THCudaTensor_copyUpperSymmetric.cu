#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
const int r = idx % n;
const int c = idx / n;
if (r > c) {
input[idx] = input[r*n + c];
}
}
}