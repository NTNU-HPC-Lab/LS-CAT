#include "hip/hip_runtime.h"
#include "includes.h"

//Problem Parameters
#define CITIES 100
#define ANTS 2000
#define MAX_DIST 100
#define ALPHA 1
#define BETA 5       //This parameter raises the weight of distance over pheromone
#define RHO 0.5      //Evapouration rate
#define QVAL 100     //
#define MAX_TOURS 50// The number of times an ant will walk trough all the cities
#define INIT_PHER (1.0/CITIES) //Initial hormone for each path
#define MAX_TOTAL_DISTANCE (CITIES * MAX_DIST) // MAX possible distance that an ant can walk


struct ant{
int curCity, nextCity, pathIndex;
int visited[CITIES];
int path[CITIES];
float tourLength;
};

//CPU
float        distances[CITIES][CITIES]; // Distance between city i an j
double       hormone[CITIES][CITIES]; //Hormone between city i and j
struct ant   ants[ANTS];
float        bestdistance[ANTS];
float 		 finalbest = (float)MAX_TOTAL_DISTANCE;
hiprandState  state[ANTS];
const size_t distances_size = sizeof(float) * size_t(CITIES*CITIES);
const size_t hormone_size = sizeof(double) * size_t(CITIES*CITIES);
//GPU
float        *distances_d;
struct ant   *ants_d;
double       *hormone_d;
float        *bestdistance_d;
hiprandState  *state_d;
int BLOCKS,THREADS;

void get_distances_matrix();
void deviceAlloc();


__global__ void initialize_ants(struct ant *ants_d, hiprandState *state_d, float *bestdistance_d , int THREADS){

int position = threadIdx.x + blockIdx.x*THREADS;
int k;

// Mark all cities as not visited
// Mark all path as not traversed
for(k = 0; k < CITIES; k++){
ants_d[position].visited[k] = 0;
ants_d[position].path[k] = -1;
}

bestdistance_d[position] = (float)MAX_TOTAL_DISTANCE;

//Random City to begin
ants_d[position].curCity = hiprand(&state_d[position])% CITIES;
ants_d[position].pathIndex = 1;
ants_d[position].path[0] = ants_d[position].curCity;
ants_d[position].nextCity = -1;
ants_d[position].tourLength = 0;
ants_d[position].visited[ants_d[position].curCity] = 1;
}