#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

// unroll 2
if (idx + blockDim.x < n)
{
g_idata[idx] += g_idata[idx + blockDim.x];
}
__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
{
if (tid < stride)
{
g_idata[idx] += g_idata[idx + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}