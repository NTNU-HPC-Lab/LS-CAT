#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling (int *g_idata, int *g_odata, unsigned int n, unsigned int q) //added int q
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * q + threadIdx.x; // q adapted idx

// unroll analogous q
if (idx + blockDim.x*(q-1) < n)
{
for (int i=1; i<q; i++)
{
g_idata[idx] += g_idata[idx + blockDim.x*i];
}
}
__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
{
if (tid < stride)
{
g_idata[idx] += g_idata[idx + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}