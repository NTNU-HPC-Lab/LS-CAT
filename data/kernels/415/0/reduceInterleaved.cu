#include "hip/hip_runtime.h"
#include "includes.h"

/*
* This code implements the interleaved Pair approaches to
* parallel reduction in CUDA. For this example, the sum operation is used.
*/

// Recursive Implementation of Interleaved Pair Approach
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// boundary check
if(idx >= n) return;

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
{
if (tid < stride)
{
g_idata[idx] += g_idata[idx + stride];
}

__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}