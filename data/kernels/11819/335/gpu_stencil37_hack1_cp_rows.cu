#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack1_cp_rows(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_CUDA_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy rows:begin\n");
printf("copy rows:gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("copy rows:blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("copy rows:tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = tile_y  * blockIdx.y;
int base_global_col   = blockDim.x*blockIdx.x;

int dst_area = n_rows*n_cols;
int s_area = gridDim.y*n_cols*2;

int base_global_idx = base_global_slice*dst_area + base_global_row * n_cols + base_global_col;

int nextRow = base_global_row+1;
bool legalNextRow = nextRow<n_rows;

int tx = threadIdx.x;
bool legalCurCol = (base_global_col + tx)<n_cols;

for(int tz=0;tz<tile_z;++tz){
bool legalCurSlice = (base_global_slice + tz)<n_slices;
int idx_dst =base_global_idx + tz*dst_area+ tx  ;
int idx = (base_global_slice+tz)*s_area + blockIdx.y*n_cols*2+blockIdx.x*blockDim.x+ tx  ;
if(legalCurCol && legalCurSlice){
shared_rows[idx] = dst[idx_dst];
}
if(legalCurCol && legalCurSlice && legalNextRow){
shared_rows[idx+n_cols] = dst[idx_dst+n_cols];
}


}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.y==0 && blockIdx.x==0 &&blockIdx.z==0 ){
if((threadIdx.x==0 || threadIdx.x==1 || threadIdx.x==2 ) && threadIdx.y==0){

int addr0 = base_global_idx+0*dst_area+threadIdx.x;
int addr  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x;
int addr1 = s_area*(base_global_slice+1)+n_cols+blockIdx.x*blockDim.x+ threadIdx.x;
int addr2 = s_area*(base_global_slice+2)+n_cols+blockIdx.x*blockDim.x+ threadIdx.x;
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst      : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,addr0,dst[addr0]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,addr,shared_rows[addr]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,addr1,shared_rows[addr1]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,addr2,shared_rows[addr2]);
}
if(threadIdx.x==0 && threadIdx.y==0){
int addr =  2*s_area+n_cols+256;
int addr1 = 2*dst_area+n_cols+256;
printf("shared_rows: addr:%d, val:%f\n", addr, shared_rows[addr]);
printf("dst        : addr:%d, val:%f\n", addr1, dst[addr1]);
}
}
#endif

#ifdef CUDA_CUDA_DEBUG

if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy rows end!\n");
}
#endif
}