#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack1_cp_cols(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0)&& threadIdx.x==0 && threadIdx.z==0){
printf("copy cols: begin\n");
printf("copy cols: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("copy cols: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("copy cols: tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = blockDim.y * blockIdx.y;
int base_global_col   = tile_x * blockIdx.x;

int area_dst = n_rows*n_cols;
int area_shared = gridDim.x*n_rows*2;

#ifdef CUDA_CUDA_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0&&threadIdx.x==0&&threadIdx.z==0)){
printf("area_shared=%d\n",area_shared);
}
#endif
int base_global_idx = base_global_slice*area_dst + base_global_row * n_cols + base_global_col;

int nextCol= base_global_col+1;
bool legalNextCol = (nextCol<n_cols)?1:0;

int ty = threadIdx.y;
bool legalCurRow = (base_global_row + ty)<n_rows;

for(int tz=0;tz<tile_z;++tz){
bool legalCurSlice = (base_global_slice + tz)<n_slices;
int idx_dst =base_global_idx + tz*area_dst + ty*n_cols ;
int idx = (base_global_slice+tz)*area_shared + blockIdx.x*2*n_rows+blockIdx.y*blockDim.y+ty;

if(legalCurRow && legalCurSlice){
shared_cols[idx] = dst[idx_dst];
}
if(legalCurRow && legalCurSlice && legalNextCol){
shared_cols[idx + n_rows] = dst[idx_dst + 1];
}

__syncthreads();
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==0 && (threadIdx.x==0)){
//		printf("shared_cols: addr:%d, val = %f\n", threadIdx.y,shared_cols[threadIdx.y]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0 && threadIdx.x==0 && threadIdx.z==0)){
printf("copy cols end!\n");
}
#endif
}