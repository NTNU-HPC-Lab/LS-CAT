#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
//Declaring the shared memory array for source
extern __shared__ double shared_mem[];
double * shSrc = shared_mem;

//indexes
int i, j;

//neighbor's values
double north, south, east, west;



//SharedMem Collumns Dimension
int smColDim = HALO*2+blockDim.y*TILE_SIZE;
int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

//Copying to shared memory

//Inner part
for ( i = 0 ; i < TILE_SIZE ; i++ )
{
for ( j = 0 ; j < TILE_SIZE ; j++ )
{
int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;
shSrc[shMemIndex]=src[globalIndex];
}
}

//Halos

if (threadIdx.x == 0 && threadIdx.y == 0 )
{

int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
//For Bottom and top row
for ( i = 0 ; i < HALO ; i++ )
{
for ( j = 0 ; j < smColDim ; j++ )
{
indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE+i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+j;
shSrc[i*smColDim+j] = src[indexTopHalo];
shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[indexBottomHalo];
}
}

//For right and left Columns
for ( i = 0 ; i < HALO ; i++ )
{
for ( j = 0 ; j < smRowDim-HALO*2; j ++ )
{
indexLeftHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + (blockIdx.y*blockDim.y*TILE_SIZE)+i;
indexRightHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE)+HALO+i;
shSrc[(HALO+j)*smColDim+i] = src[indexLeftHalo];
shSrc[(HALO+j+1)*smColDim-HALO+i] = src[indexRightHalo];
}
}
}

__syncthreads();



for ( i = 0 ; i < TILE_SIZE ; i++ )
{
for ( j = 0 ; j < TILE_SIZE ; j++ )
{
int globalIndex=HALO*N+blockIdx.x*blockDim.x*TILE_SIZE*N+threadIdx.x*TILE_SIZE*N+i*N+blockIdx.y*blockDim.y*TILE_SIZE+threadIdx.y*TILE_SIZE+j+HALO;
int shMemIndex=HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;


//Getting the neighbohrs
north = shSrc[shMemIndex-smColDim];
south = shSrc[shMemIndex+smColDim];
east  = shSrc[shMemIndex+1];
west  = shSrc[shMemIndex-1];
//Real Stencil operation
dst[globalIndex] = ( north + south + east + west )/5.5;
}
}

__syncthreads();
}