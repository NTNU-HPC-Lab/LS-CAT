#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack1_cp_slices(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,uint64_t n_rows, uint64_t n_cols,uint64_t n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices begin!\n");
printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = tile_y * blockIdx.y;
int base_global_col   = blockDim.x * blockIdx.x;

uint64_t area = n_rows*n_cols;
uint64_t base_global_idx = base_global_slice*area + base_global_row * n_cols + base_global_col;

int nextSlice = base_global_slice+1;
bool legalNextSlice = (nextSlice<n_slices);
int tx = threadIdx.x;
bool legalCurCol = (base_global_col + tx)<n_cols;

for(int ty=0;ty<tile_y;++ty){
bool legalCurRow = (base_global_row + ty)<n_rows;
uint64_t idx = blockIdx.z*area*2 + (base_global_row+ty)*n_cols + base_global_col+tx ;
uint64_t idx_dst = base_global_idx + ty*n_cols+tx;
if(legalCurCol&&legalCurRow){
shared_slices[idx] = dst[idx_dst];
}
if(legalNextSlice&&legalCurCol&&legalCurRow){
shared_slices[idx+area] = dst[idx_dst+area];
}

}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==1 ){
//	printf("shared_slices: addr:%d, val = %f\n",n_cols*n_rows + threadIdx.x,shared_slices[n_cols*n_rows+threadIdx.x]);
if(threadIdx.x==0||threadIdx.x==1||threadIdx.x==2){
int addr = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x;
int addr1 = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x+n_cols;
int addr2 = n_cols*n_rows + blockDim.x*blockIdx.x+threadIdx.x+n_cols*2;
printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr,shared_slices[addr]);
printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr1,shared_slices[addr1]);
printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr2,shared_slices[addr2]);
}
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices end!\n");
}
#endif
}