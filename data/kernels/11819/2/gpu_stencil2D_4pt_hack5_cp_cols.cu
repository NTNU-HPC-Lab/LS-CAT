#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
* GPU Device kernel for the for 2D stencil
* First attempt during hackaton
* M = Rows, N = Cols INCLUDING HALOS
* In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows
*/

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows,int tile_x,int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols begin!\n");
}
#endif

int base_global_row = tile_y  * blockIdx.y;
int base_global_col = tile_x  * blockIdx.x;
int base_global_idx = N*base_global_row + base_global_col ;
int nextCol = base_global_col+1;
bool legalNextCol = (nextCol<N);
int t = threadIdx.y;
int idx = 2*M*blockIdx.x + t + base_global_row;
int idx_nextCol = idx + M ;
bool legalCurRow = (base_global_row + t)<M;
if(legalCurRow){
shared_cols[idx] = dst[base_global_idx + t*N];
}
if(legalNextCol && legalCurRow){
shared_cols[idx_nextCol] = dst[base_global_idx + t*N+1];
}
__syncthreads();


#ifdef CUDA_CUDA_DEBUG
//	if(threadIdx.y==0){
//		printf("blockDimy = %d\n",blockDim.y);
//	}
if(blockIdx.x==1 && t<5){
printf("addr: %d ,%f,\n",idx_nextCol,shared_cols[idx_nextCol]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols finish!\n");
}
#endif
}