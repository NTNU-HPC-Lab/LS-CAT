#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){


#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
printf("copy rows begin!\n");
}
#endif

int base_global_row = (tile_y  * blockIdx.y );
int base_global_col = blockDim.x*blockIdx.x;
int base_global_idx = N*base_global_row + base_global_col ;
int nextRow = base_global_row+1;
bool legalNextRow = (nextRow<M)?1:0;
int t = threadIdx.x;
bool legalCurCol = (base_global_col + t)<N;
int idx = (base_global_row/tile_y)*2*N + t+base_global_col;
int idx_nextrow = idx + N;
if(legalCurCol){
shared_rows[idx] = dst[base_global_idx + t];
}
if(legalNextRow&&legalCurCol){
shared_rows[idx_nextrow] = dst[base_global_idx + N+t];
}
__syncthreads();


#ifdef CUDA_DARTS_DEBUG
//	if(threadIdx.x==0){
//		printf("blockIdx.x = %d,blockIdx.y = %d\n",blockIdx.x,blockIdx.y);
//	}
//	if(blockIdx.y==1 && threadIdx.x==0){
//		printf("addr: %d\n",idx_nextrow);
//	}
if(blockIdx.y==0 && blockIdx.x==2 && (t==0 || t==1)){
printf("addr:%d, val = %f\n", idx_nextrow,shared_rows[idx_nextrow]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
printf("copy rows finish!\n");
}
#endif
}