#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack2_cp_cols(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int d_xpitch, int d_ypitch, int d_zpitch,int s_xpitch,int s_ypitch, int s_zpitch, int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_CUDA_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0)&& threadIdx.x==0 && threadIdx.z==0){
printf("copy cols: begin\n");
printf("copy cols: n_cols=%d,n_rows=%d,n_slices=%d\n",n_cols,n_rows,n_slices);
printf("copy cols: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("copy cols: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("copy cols: tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = blockDim.y * blockIdx.y;
int base_global_col   = tile_x * blockIdx.x;

//int dst_area = n_rows*n_cols;
//int shared_area = gridDim.x*n_rows*2;
int dst_area = d_ypitch*d_xpitch;
int shared_area = gridDim.x*s_ypitch*2;

#ifdef CUDA_CUDA_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0&&threadIdx.x==0&&threadIdx.z==0)){
printf("copy cols: shared_area=%d\n",shared_area);
}
#endif
//int base_global_idx = base_global_slice*dst_area + base_global_row * n_cols + base_global_col;
int base_global_idx = base_global_slice*dst_area + base_global_row * d_xpitch + base_global_col;

int nextCol= base_global_col+1;
bool legalNextCol = (nextCol<n_cols)?1:0;

int ty = threadIdx.y;
bool legalCurRow = (base_global_row + ty)<n_rows;

for(int tz=0;tz<tile_z;++tz){
bool legalCurSlice = (base_global_slice + tz)<n_slices;
//int idx = (base_global_slice+tz)*shared_area + blockIdx.x*2*n_rows+blockIdx.y*blockDim.y+ty;
//int idx_dst =base_global_idx + tz*dst_area + ty*n_cols ;
int idx = (base_global_slice+tz)*shared_area + blockIdx.x*2*s_ypitch+blockIdx.y*blockDim.y+ty;
int idx_dst =base_global_idx + tz*dst_area + ty*d_xpitch ;

if(legalCurRow && legalCurSlice){
shared_cols[idx] = dst[idx_dst];
}
if(legalCurRow && legalCurSlice && legalNextCol){
//shared_cols[idx + n_rows] = dst[idx_dst + 1];
shared_cols[idx + s_ypitch] = dst[idx_dst + 1];
}

__syncthreads();
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==0 && (threadIdx.x==0)){
//		printf("shared_cols: addr:%d, val = %f\n", threadIdx.y,shared_cols[threadIdx.y]);
}

if(blockIdx.y==0 && blockIdx.x==0 &&blockIdx.z==0 ){
if((threadIdx.x==0 || threadIdx.x==1 || threadIdx.x==2 ) && threadIdx.y==0){

int d_addr0 = base_global_idx+0*dst_area+threadIdx.x;
int d_addr1 = base_global_idx+1*dst_area+threadIdx.x;
int addr  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x;
int addr1 = shared_area*(base_global_slice+1)+blockIdx.x*blockDim.x+ threadIdx.x;
int addr2 = shared_area*(base_global_slice+2)+blockIdx.x*blockDim.x+ threadIdx.x;
printf("copy cols: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst        : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,d_addr0,dst[d_addr0]);
printf("copy cols: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst        : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,d_addr1,dst[d_addr1]);
printf("copy cols: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_cols: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,addr,shared_cols[addr]);
printf("copy cols: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_cols: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,addr1,shared_cols[addr1]);
printf("copy cols: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_cols: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,addr2,shared_cols[addr2]);
}
}

#endif

#ifdef CUDA_CUDA_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0 && threadIdx.x==0 && threadIdx.z==0)){
printf("copy cols end!\n");
}
#endif
}