#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
extern __shared__ int shared_mem[];
int * shmem=shared_mem;
shmem[threadIdx.x]=threadIdx.x;
a[threadIdx.x]=shmem[threadIdx.x];
b[threadIdx.x]=shmem[threadIdx.x];
c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x];
}