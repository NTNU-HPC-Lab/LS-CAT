#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
* GPU Device kernel for the for 2D stencil
* First attempt during hackaton
* M = Rows, N = Cols INCLUDING HALOS
* In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows
*/

__global__ void gpu_stencil2D_4pt_hack2(double * dst, double * src, int M, int N)
{
//	printf("kernel begin!\n");
//Declaring the shared memory array for source
__shared__ double shared_mem[ 1 + HALO*2 ] [ GRID_TILE_X + HALO*2]; //1 is the row I am modifying
//double * shSrc = shared_mem;

//indexes
int i, j, curRow;
//Cols   *  numRows/Tile * tileIndex
int base_global_idx = ( N ) * ( GRID_TILE_Y * blockIdx.y ) + GRID_TILE_X*blockIdx.x;

int center = 1,north = 0,south = 2; //indexes for the current location in the shared memory

//copy the shared memory to fill the pipeline
for (i = 0 ; i < 1+HALO*2 ; i ++ )
for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
{
shared_mem [i][j] = src[base_global_idx + i*N + j];
}
__syncthreads();
//Pipelined copy one row and process it
for ( curRow = HALO; curRow < GRID_TILE_Y; curRow+=1 )
{
//Stencil computation
for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
{
//top             + bottom              + left                + right
dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
}

__syncthreads();
//We are copying from dst to shared memory.
for (j = threadIdx.x ; j < GRID_TILE_X+2*HALO ; j+=blockDim.x)
{
shared_mem [north][j] = src[base_global_idx + (curRow+2)*N + j];
}

center = ROTATE_UP(center,3);
south  = ROTATE_UP(south,3);
north = ROTATE_UP(north,3);
__syncthreads();
}

//Dranning the pipeline
for (j = threadIdx.x + HALO ; j < GRID_TILE_X+HALO ; j+=blockDim.x)
{
//top             + bottom              + left                + right
dst[base_global_idx + curRow*N + j] = (shared_mem[north][j] + shared_mem[south][j] + shared_mem[center][j-1] + shared_mem[center][j+1] )/5.5;
}
__syncthreads();

//	printf("kernel finish!\n");
}