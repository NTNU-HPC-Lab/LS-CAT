#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack2_cp_slices(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int d_xpitch,int d_ypitch,int d_zpitch,int s_xpitch,int s_ypitch, int s_zpitch, int n_rows, int n_cols,int n_slices, int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices: begin!\n");
printf("copy slices: n_cols=%d,n_rows=%d,n_slices=%d\n",n_cols,n_rows,n_slices);
printf("copy slices: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("copy slices: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("copy slices: tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = tile_y * blockIdx.y;
int base_global_col   = blockDim.x * blockIdx.x;

//int area = n_rows*n_cols;
//int base_global_idx = base_global_slice*area + base_global_row * n_cols + base_global_col;
//int d_area = n_rows*d_xpitch;
//int s_area = n_rows*n_cols;
int d_area = d_ypitch*d_xpitch;
int s_area = s_ypitch*s_xpitch;
int base_global_idx = base_global_slice*d_area + base_global_row * d_xpitch + base_global_col;

int nextSlice = base_global_slice+1;
bool legalNextSlice = (nextSlice<n_slices);
int tx = threadIdx.x;
bool legalCurCol = (base_global_col + tx)<n_cols;

for(int ty=0;ty<tile_y;++ty){
bool legalCurRow = (base_global_row + ty)<n_rows;
//int s_idx = blockIdx.z*s_area*2 + (base_global_row+ty)*n_cols + base_global_col+tx ;
//int dst_idx = base_global_idx + ty*n_cols+tx;
int s_idx = blockIdx.z*s_area*2 + (base_global_row+ty)*s_xpitch + base_global_col+tx ;
int d_idx = base_global_idx + ty*d_xpitch+tx;
if(legalCurCol&&legalCurRow){
shared_slices[s_idx] = dst[d_idx];
}
if(legalNextSlice&&legalCurCol&&legalCurRow){
shared_slices[s_idx+s_area] = dst[d_idx+d_area];
}

}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==0 ){
//	printf("shared_slices: addr:%d, val = %f\n",n_cols*n_rows + threadIdx.x,shared_slices[n_cols*n_rows+threadIdx.x]);
if(threadIdx.x==0||threadIdx.x==1||threadIdx.x==2){
int addr  = s_xpitch*s_ypitch + blockDim.x*blockIdx.x+threadIdx.x;
int addr1 = s_xpitch*s_ypitch + blockDim.x*blockIdx.x+threadIdx.x+s_xpitch;
int addr2 = s_xpitch*s_ypitch + blockDim.x*blockIdx.x+threadIdx.x+s_xpitch*2;

int daddr  = d_xpitch*d_ypitch + blockDim.x*blockIdx.x+threadIdx.x;
int daddr1 = d_xpitch*d_ypitch + blockDim.x*blockIdx.x+threadIdx.x+d_xpitch;
int daddr2 = d_xpitch*d_ypitch + blockDim.x*blockIdx.x+threadIdx.x+d_xpitch*2;
printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, daddr,dst[daddr]);
printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, daddr1,dst[daddr1]);
printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, daddr2,dst[daddr2]);

printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr,shared_slices[addr]);
printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr1,shared_slices[addr1]);
printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",blockIdx.x, blockIdx.y, blockIdx.z, addr2,shared_slices[addr2]);
}
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices end!\n");
}
#endif
}