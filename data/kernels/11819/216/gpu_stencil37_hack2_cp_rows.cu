#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack2_cp_rows(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int d_xpitch,int d_ypitch,int d_zpitch,int s_xpitch,int s_ypitch, int s_zpitch, int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy rows: begin\n");
printf("copy rows: n_cols=%d,n_rows=%d,n_slices=%d\n",n_cols,n_rows,n_slices);
printf("copy rows: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("copy rows: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("copy rows: tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif
int base_global_slice = tile_z * blockIdx.z;
int base_global_row   = tile_y  * blockIdx.y;
int base_global_col   = blockDim.x*blockIdx.x;

//int dst_area = n_rows*n_cols;
//int s_area = gridDim.y*n_cols*2;
int dst_area = d_ypitch*d_xpitch;
int s_area = gridDim.y*s_xpitch*2;

//int base_global_idx = base_global_slice*dst_area + base_global_row * n_cols + base_global_col;
int base_global_idx = base_global_slice*dst_area + base_global_row * d_xpitch + base_global_col;

int nextRow = base_global_row+1;
bool legalNextRow = nextRow<n_rows;

int tx = threadIdx.x;
bool legalCurCol = (base_global_col + tx)<n_cols;

for(int tz=0;tz<tile_z;++tz){
bool legalCurSlice = (base_global_slice + tz)<n_slices;
int idx_dst =base_global_idx + tz*dst_area+ tx  ;
//int idx = (base_global_slice+tz)*s_area + blockIdx.y*n_cols*2+blockIdx.x*blockDim.x+ tx  ;
int idx = (base_global_slice+tz)*s_area + blockIdx.y*s_xpitch*2+blockIdx.x*blockDim.x+ tx  ;
if(legalCurCol && legalCurSlice){
shared_rows[idx] = dst[idx_dst];
}
if(legalCurCol && legalCurSlice && legalNextRow){
//shared_rows[idx+n_cols] = dst[idx_dst+n_cols];
shared_rows[idx+s_xpitch] = dst[idx_dst+d_xpitch];
}


}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.y==0 && blockIdx.x==0 &&blockIdx.z==0 ){
if((threadIdx.x==0 || threadIdx.x==1 || threadIdx.x==2 ) && threadIdx.y==0){

int d_addr0 = base_global_idx+0*dst_area+threadIdx.x;
int d_addr1 = base_global_idx+1*dst_area+threadIdx.x;
int s_addr00  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x;
int s_addr01  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x+s_xpitch;
int s_addr02  = base_global_slice+blockIdx.x*blockDim.x + threadIdx.x+s_xpitch*2;
int s_addr10 = s_area*(base_global_slice+1)+blockIdx.x*blockDim.x+ threadIdx.x;
int s_addr11 = s_area*(base_global_slice+1)+blockIdx.x*blockDim.x+ threadIdx.x+s_xpitch;
int s_addr12 = s_area*(base_global_slice+1)+blockIdx.x*blockDim.x+ threadIdx.x+s_xpitch*2;
int s_addr20 = s_area*(base_global_slice+2)+blockIdx.x*blockDim.x+ threadIdx.x;
int s_addr21 = s_area*(base_global_slice+2)+blockIdx.x*blockDim.x+ threadIdx.x+s_xpitch;
int s_addr22 = s_area*(base_global_slice+2)+blockIdx.x*blockDim.x+ threadIdx.x+s_xpitch*2;
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst        : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,d_addr0,dst[d_addr0]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,dst        : z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,d_addr1,dst[d_addr1]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,0,s_addr00,shared_rows[s_addr00]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,s_addr01,shared_rows[s_addr01]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,s_addr00,shared_rows[s_addr02]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,s_addr10,shared_rows[s_addr10]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,s_addr11,shared_rows[s_addr11]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,1,s_addr12,shared_rows[s_addr12]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,s_addr20,shared_rows[s_addr20]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,s_addr21,shared_rows[s_addr21]);
printf("copy rows: blockIdx.x=%d, blockIdx.y=%d,blockIdx.z=%d,shared_rows: z:%d, addr:%d, val = %f\n",blockIdx.x, blockIdx.y,blockIdx.z,2,s_addr22,shared_rows[s_addr22]);
}
if(threadIdx.x==0 && threadIdx.y==0){
int addr =  2*s_area+n_cols+256;
int addr1 = 2*dst_area+n_cols+256;
printf("copy rows: shared_rows: addr:%d, val:%f\n", addr, shared_rows[addr]);
printf("copy rows: dst        : addr:%d, val:%f\n", addr1, dst[addr1]);
}
}
#endif

#ifdef CUDA_DARTS_DEBUG

if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy rows end!\n");
}
#endif
}