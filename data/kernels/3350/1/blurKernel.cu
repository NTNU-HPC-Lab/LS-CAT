#include "hip/hip_runtime.h"
#include "includes.h"


#define index(i, j, w)  ((i)*(w)) + (j)


__global__ void blurKernel (unsigned char * d_inputArray, unsigned char * d_outputArray, int w, int h, int blurSize){

int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

if(Col<w && Row < h){
int pixVal = 0;
int pixels = 0;

for(int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow){
for(int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol){
int curRow = Row + blurRow;
int curCol = Col + blurCol;

//verify we have a valid image pixel
if(curRow > -1 && curRow < h && curCol > -1 && curCol < w){
pixVal += d_inputArray[curRow*w+curCol];
pixels++; // keep track of number of pixels in the avg
}
}
}

//write our new pixel value out
d_outputArray[Row*w+Col] = (unsigned char)(pixVal/pixels);


}

}