#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_for_neuron_spikes_kernel(float *d_membrane_potentials_v, float *d_thresholds_for_action_potential_spikes, float *d_resting_potentials, float* d_last_spike_time_of_each_neuron, unsigned char* d_bitarray_of_neuron_spikes, int bitarray_length, int bitarray_maximum_axonal_delay_in_timesteps, float current_time_in_seconds, float timestep, size_t total_number_of_neurons, bool high_fidelity_spike_flag) {

// Get thread IDs
int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {
if (d_membrane_potentials_v[idx] >= d_thresholds_for_action_potential_spikes[idx]) {

// Set current time as last spike time of neuron
d_last_spike_time_of_each_neuron[idx] = current_time_in_seconds;

// Reset membrane potential
d_membrane_potentials_v[idx] = d_resting_potentials[idx];

// High fidelity spike storage
if (high_fidelity_spike_flag){
// Get start of the given neuron's bits
int neuron_id_spike_store_start = idx * bitarray_length;
// Get offset depending upon the current timestep
int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
int offset_byte = offset_index / 8;
int offset_bit_pos = offset_index - (8 * offset_byte);
// Get the specific position at which we should be putting the current value
unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
// Set the specific bit in the byte to on
byte |= (1 << offset_bit_pos);
// Assign the byte
d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
}

} else {
// High fidelity spike storage
if (high_fidelity_spike_flag){
// Get start of the given neuron's bits
int neuron_id_spike_store_start = idx * bitarray_length;
// Get offset depending upon the current timestep
int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
int offset_byte = offset_index / 8;
int offset_bit_pos = offset_index - (8 * offset_byte);
// Get the specific position at which we should be putting the current value
unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
// Set the specific bit in the byte to on
byte &= ~(1 << offset_bit_pos);
// Assign the byte
d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
}
}

idx += blockDim.x * gridDim.x;
}
__syncthreads();

}