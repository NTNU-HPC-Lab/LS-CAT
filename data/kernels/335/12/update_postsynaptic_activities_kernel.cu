#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_postsynaptic_activities_kernel( float timestep, size_t total_number_of_neurons, float * d_recent_postsynaptic_activities_D, float * d_last_spike_time_of_each_neuron, float current_time_in_seconds, float decay_term_tau_D, float model_parameter_alpha_D) {

int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {

// if (d_stdp[idx] == 1) {

float recent_postsynaptic_activity_D = d_recent_postsynaptic_activities_D[idx];

float new_recent_postsynaptic_activity_D = (1 - (timestep/decay_term_tau_D)) * recent_postsynaptic_activity_D;

if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
new_recent_postsynaptic_activity_D += timestep * model_parameter_alpha_D * (1 - recent_postsynaptic_activity_D);
}

d_recent_postsynaptic_activities_D[idx] = new_recent_postsynaptic_activity_D;

// }

idx += blockDim.x * gridDim.x;

}
}