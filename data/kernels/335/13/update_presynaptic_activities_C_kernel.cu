#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_presynaptic_activities_C_kernel(float* d_recent_presynaptic_activities_C, float* d_time_of_last_spike_to_reach_synapse, bool* d_stdp, float timestep, float current_time_in_seconds, size_t total_number_of_synapses, float synaptic_neurotransmitter_concentration_alpha_C, float decay_term_tau_C) {

int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
int idx = t_idx;
while (idx < total_number_of_synapses) {

if (d_stdp[idx] == true) {

float recent_presynaptic_activity_C = d_recent_presynaptic_activities_C[idx];

float new_recent_presynaptic_activity_C = (1 - (timestep/decay_term_tau_C)) * recent_presynaptic_activity_C;

if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
new_recent_presynaptic_activity_C += timestep * synaptic_neurotransmitter_concentration_alpha_C * (1 - recent_presynaptic_activity_C);
}

if (recent_presynaptic_activity_C != new_recent_presynaptic_activity_C) {
d_recent_presynaptic_activities_C[idx] = new_recent_presynaptic_activity_C;
}

}

idx += blockDim.x * gridDim.x;

}

}