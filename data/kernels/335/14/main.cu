#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "update_synaptic_efficacies_or_weights_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_recent_presynaptic_activities_C = NULL;
hipMalloc(&d_recent_presynaptic_activities_C, XSIZE*YSIZE);
float *d_recent_postsynaptic_activities_D = NULL;
hipMalloc(&d_recent_postsynaptic_activities_D, XSIZE*YSIZE);
int *d_postsynaptic_neuron_indices = NULL;
hipMalloc(&d_postsynaptic_neuron_indices, XSIZE*YSIZE);
float *d_synaptic_efficacies_or_weights = NULL;
hipMalloc(&d_synaptic_efficacies_or_weights, XSIZE*YSIZE);
float current_time_in_seconds = 1;
float *d_time_of_last_spike_to_reach_synapse = NULL;
hipMalloc(&d_time_of_last_spike_to_reach_synapse, XSIZE*YSIZE);
float *d_last_spike_time_of_each_neuron = NULL;
hipMalloc(&d_last_spike_time_of_each_neuron, XSIZE*YSIZE);
bool *d_stdp = NULL;
hipMalloc(&d_stdp, XSIZE*YSIZE);
size_t total_number_of_synapses = 1;
float learning_rate_rho = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
update_synaptic_efficacies_or_weights_kernel<<<gridBlock,threadBlock>>>(d_recent_presynaptic_activities_C,d_recent_postsynaptic_activities_D,d_postsynaptic_neuron_indices,d_synaptic_efficacies_or_weights,current_time_in_seconds,d_time_of_last_spike_to_reach_synapse,d_last_spike_time_of_each_neuron,d_stdp,total_number_of_synapses,learning_rate_rho);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
update_synaptic_efficacies_or_weights_kernel<<<gridBlock,threadBlock>>>(d_recent_presynaptic_activities_C,d_recent_postsynaptic_activities_D,d_postsynaptic_neuron_indices,d_synaptic_efficacies_or_weights,current_time_in_seconds,d_time_of_last_spike_to_reach_synapse,d_last_spike_time_of_each_neuron,d_stdp,total_number_of_synapses,learning_rate_rho);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
update_synaptic_efficacies_or_weights_kernel<<<gridBlock,threadBlock>>>(d_recent_presynaptic_activities_C,d_recent_postsynaptic_activities_D,d_postsynaptic_neuron_indices,d_synaptic_efficacies_or_weights,current_time_in_seconds,d_time_of_last_spike_to_reach_synapse,d_last_spike_time_of_each_neuron,d_stdp,total_number_of_synapses,learning_rate_rho);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}