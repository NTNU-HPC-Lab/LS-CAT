#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_random_states_kernel(unsigned int seed, hiprandState_t* d_states, size_t total_number) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
// int idx_g = idx;
if (idx < total_number) {
hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
idx, /* the sequence number should be different for each core (unless you want all
cores to get the same sequence of numbers for some reason - use thread id! */
0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&d_states[idx]);

__syncthreads();
}
}