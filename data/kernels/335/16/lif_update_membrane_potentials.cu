#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lif_update_membrane_potentials(float *d_membrane_potentials_v, float * d_membrane_resistances_R, float * d_membrane_time_constants_tau_m, float * d_resting_potentials, float* d_current_injections, float timestep, size_t total_number_of_neurons){


// // Get thread IDs
int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {

float equation_constant = timestep / d_membrane_time_constants_tau_m[idx];
float membrane_potential_Vi = d_membrane_potentials_v[idx];
float current_injection_Ii = d_current_injections[idx];
float resting_potential_V0 = d_resting_potentials[idx];
float temp_membrane_resistance_R = d_membrane_resistances_R[idx];

float new_membrane_potential = equation_constant * (resting_potential_V0 + temp_membrane_resistance_R * current_injection_Ii) + (1 - equation_constant) * membrane_potential_Vi;

d_membrane_potentials_v[idx] = new_membrane_potential;

idx += blockDim.x * gridDim.x;

}
__syncthreads();
}