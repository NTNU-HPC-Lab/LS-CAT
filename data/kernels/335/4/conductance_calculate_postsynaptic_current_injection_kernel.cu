#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conductance_calculate_postsynaptic_current_injection_kernel(int * d_presynaptic_neuron_indices, int* d_postsynaptic_neuron_indices, float* d_reversal_potentials_Vhat, float* d_neurons_current_injections, size_t total_number_of_synapses, float * d_membrane_potentials_v, float * d_synaptic_conductances_g){

int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_synapses) {

float reversal_potential_Vhat = d_reversal_potentials_Vhat[idx];
int postsynaptic_neuron_index = d_postsynaptic_neuron_indices[idx];
float membrane_potential_v = d_membrane_potentials_v[postsynaptic_neuron_index];
float synaptic_conductance_g = d_synaptic_conductances_g[idx];

float component_for_sum = synaptic_conductance_g * (reversal_potential_Vhat - membrane_potential_v);
if (component_for_sum != 0.0) {
atomicAdd(&d_neurons_current_injections[postsynaptic_neuron_index], component_for_sum);
}

idx += blockDim.x * gridDim.x;

}
__syncthreads();
}