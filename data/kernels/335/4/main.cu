#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "conductance_calculate_postsynaptic_current_injection_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *d_presynaptic_neuron_indices = NULL;
hipMalloc(&d_presynaptic_neuron_indices, XSIZE*YSIZE);
int *d_postsynaptic_neuron_indices = NULL;
hipMalloc(&d_postsynaptic_neuron_indices, XSIZE*YSIZE);
float *d_reversal_potentials_Vhat = NULL;
hipMalloc(&d_reversal_potentials_Vhat, XSIZE*YSIZE);
float *d_neurons_current_injections = NULL;
hipMalloc(&d_neurons_current_injections, XSIZE*YSIZE);
size_t total_number_of_synapses = 1;
float *d_membrane_potentials_v = NULL;
hipMalloc(&d_membrane_potentials_v, XSIZE*YSIZE);
float *d_synaptic_conductances_g = NULL;
hipMalloc(&d_synaptic_conductances_g, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(d_presynaptic_neuron_indices,d_postsynaptic_neuron_indices,d_reversal_potentials_Vhat,d_neurons_current_injections,total_number_of_synapses,d_membrane_potentials_v,d_synaptic_conductances_g);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(d_presynaptic_neuron_indices,d_postsynaptic_neuron_indices,d_reversal_potentials_Vhat,d_neurons_current_injections,total_number_of_synapses,d_membrane_potentials_v,d_synaptic_conductances_g);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(d_presynaptic_neuron_indices,d_postsynaptic_neuron_indices,d_reversal_potentials_Vhat,d_neurons_current_injections,total_number_of_synapses,d_membrane_potentials_v,d_synaptic_conductances_g);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}