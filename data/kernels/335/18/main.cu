#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "reset_states_u_after_spikes_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_states_u = NULL;
hipMalloc(&d_states_u, XSIZE*YSIZE);
float *d_param_d = NULL;
hipMalloc(&d_param_d, XSIZE*YSIZE);
float *d_last_spike_time_of_each_neuron = NULL;
hipMalloc(&d_last_spike_time_of_each_neuron, XSIZE*YSIZE);
float current_time_in_seconds = 1;
size_t total_number_of_neurons = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
reset_states_u_after_spikes_kernel<<<gridBlock,threadBlock>>>(d_states_u,d_param_d,d_last_spike_time_of_each_neuron,current_time_in_seconds,total_number_of_neurons);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
reset_states_u_after_spikes_kernel<<<gridBlock,threadBlock>>>(d_states_u,d_param_d,d_last_spike_time_of_each_neuron,current_time_in_seconds,total_number_of_neurons);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
reset_states_u_after_spikes_kernel<<<gridBlock,threadBlock>>>(d_states_u,d_param_d,d_last_spike_time_of_each_neuron,current_time_in_seconds,total_number_of_neurons);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}