#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reset_states_u_after_spikes_kernel(float *d_states_u, float * d_param_d, float* d_last_spike_time_of_each_neuron, float current_time_in_seconds, size_t total_number_of_neurons) {

int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {
if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {

d_states_u[idx] += d_param_d[idx];

}
idx += blockDim.x * gridDim.x;
}
__syncthreads();
}