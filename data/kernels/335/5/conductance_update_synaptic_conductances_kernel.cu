#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conductance_update_synaptic_conductances_kernel(float timestep, float * d_synaptic_conductances_g, float * d_synaptic_efficacies_or_weights, float * d_time_of_last_spike_to_reach_synapse, float * d_biological_conductance_scaling_constants_lambda, int total_number_of_synapses, float current_time_in_seconds, float * d_decay_terms_tau_g) {

int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_synapses) {

float synaptic_conductance_g = d_synaptic_conductances_g[idx];

float new_conductance = (1.0 - (timestep/d_decay_terms_tau_g[idx])) * synaptic_conductance_g;

if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
float timestep_times_synaptic_efficacy = timestep * d_synaptic_efficacies_or_weights[idx];
float biological_conductance_scaling_constant_lambda = d_biological_conductance_scaling_constants_lambda[idx];
float timestep_times_synaptic_efficacy_times_scaling_constant = timestep_times_synaptic_efficacy * biological_conductance_scaling_constant_lambda;
new_conductance += timestep_times_synaptic_efficacy_times_scaling_constant;
}

if (synaptic_conductance_g != new_conductance) {
d_synaptic_conductances_g[idx] = new_conductance;
}

idx += blockDim.x * gridDim.x;
}
__syncthreads();

}