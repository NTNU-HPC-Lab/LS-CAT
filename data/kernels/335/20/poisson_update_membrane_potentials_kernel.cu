#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states, float *d_rates, float *d_membrane_potentials_v, float timestep, float * d_thresholds_for_action_potential_spikes, size_t total_number_of_input_neurons, int current_stimulus_index) {


int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
int idx = t_idx;
while (idx < total_number_of_input_neurons){

int rate_index = (total_number_of_input_neurons * current_stimulus_index) + idx;

float rate = d_rates[rate_index];

if (rate > 0.1) {

// Creates random float between 0 and 1 from uniform distribution
// d_states effectively provides a different seed for each thread
// hiprand_uniform produces different float every time you call it
float random_float = hiprand_uniform(&d_states[t_idx]);

// if the randomnumber is less than the rate
if (random_float < (rate * timestep)) {

// Puts membrane potential above default spiking threshold
d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;

}

}

idx += blockDim.x * gridDim.x;

}
__syncthreads();
}