#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_for_generator_spikes_kernel(int *d_neuron_ids_for_stimulus, float *d_spike_times_for_stimulus, float* d_last_spike_time_of_each_neuron, unsigned char* d_bitarray_of_neuron_spikes, int bitarray_length, int bitarray_maximum_axonal_delay_in_timesteps, float current_time_in_seconds, float timestep, size_t number_of_spikes_in_stimulus, bool high_fidelity_spike_flag) {

// // Get thread IDs
int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < number_of_spikes_in_stimulus) {
if (fabs(current_time_in_seconds - d_spike_times_for_stimulus[idx]) < 0.5 * timestep) {
__syncthreads();
d_last_spike_time_of_each_neuron[d_neuron_ids_for_stimulus[idx]] = current_time_in_seconds;

if (high_fidelity_spike_flag){
// Get start of the given neuron's bits
int neuron_id_spike_store_start = d_neuron_ids_for_stimulus[idx] * bitarray_length;
// Get offset depending upon the current timestep
int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
int offset_byte = offset_index / 8;
int offset_bit_pos = offset_index - (8 * offset_byte);
// Get the specific position at which we should be putting the current value
unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
// Set the specific bit in the byte to on
byte |= (1 << offset_bit_pos);
// Assign the byte
d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
}
} else {
// High fidelity spike storage
if (high_fidelity_spike_flag){
// Get start of the given neuron's bits
int neuron_id_spike_store_start = d_neuron_ids_for_stimulus[idx] * bitarray_length;
// Get offset depending upon the current timestep
int offset_index = (int)(round((float)(current_time_in_seconds / timestep))) % bitarray_maximum_axonal_delay_in_timesteps;
int offset_byte = offset_index / 8;
int offset_bit_pos = offset_index - (8 * offset_byte);
// Get the specific position at which we should be putting the current value
unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
// Set the specific bit in the byte to on
byte &= ~(1 << offset_bit_pos);
// Assign the byte
d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte] = byte;
}
}

idx += blockDim.x * gridDim.x;
}
__syncthreads();
}