#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


#define n 10
#define W 100

hipError_t knapsackCuda(int *output, const int *val, const int *wt, unsigned int size);

__device__ int maxi(int a, int b) {
return (a > b)? a : b;
}
__global__ void knapsackKernel(int *wt, int *val, int *output, int i) {
int w = threadIdx.x;

//__syncthreads();
if (i == 0 || w == 0)
output[(i*W)+w] = 0;
else if (wt[i-1] <= w)
output[(i*W)+w] = maxi(val[i-1] + output[((i-1)*W)+(w-wt[i-1])],  output[((i-1)*W)+w]);
else
output[(i*W)+w] = output[((i-1)*W)+w];
__syncthreads();

}