#include "hip/hip_runtime.h"
#include "includes.h"
// a simple CUDA kernel to add two vectors

extern "C"
{


} // extern "C"

__global__ void vadd(const float *a, const float *b, float *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = a[i] + b[i];
}