#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
int first = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

hiprandState local_state = state[first];
for (int id = first ; id < n ; id += stride) {
destinations[id] = sources[hiprand(&local_state) % n];
}

state[first] = local_state;
}