#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(hipSurfaceObject_t input, hipSurfaceObject_t output, int width, int height) {

//Get the pixel index
unsigned int xPx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int yPx = threadIdx.y + blockIdx.y * blockDim.y;


//Don't do any computation if this thread is outside of the surface bounds.
if (xPx >= width || yPx >= height) return;

//Copy the contents of input to output.
uchar4 pixel = { 255,128,0,255 };
//Read a pixel from the input. Disable to default to the flat orange color above
surf2Dread<uchar4>(&pixel, input, xPx * sizeof(uchar4), yPx, hipBoundaryModeClamp);

//Invert the color
pixel.x = ~pixel.x;
pixel.y = ~pixel.y;
pixel.z = ~pixel.z;

//Write the new pixel color to the
surf2Dwrite(pixel, output, xPx * sizeof(uchar4), yPx);
}