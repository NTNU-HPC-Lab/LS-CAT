#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *array_a, float *array_b, float *array_c, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int step = blockDim.x * gridDim.x;

for (int i = tid; i < size; i += step) {
array_c[i] = array_a[i] + array_b[i];
}
}