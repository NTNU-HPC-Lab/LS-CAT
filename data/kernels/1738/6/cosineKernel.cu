#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cosineKernel(float *a, float *b, float *outN, float *outD1, float *outD2, int size) {
extern __shared__ float sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
int stride = gridDim.x * blockDim.x;
while (i < size) {
sdata[3 * tid] = a[i] * b[i] + a[i + blockDim.x] * b[i + blockDim.x];
sdata[3 * tid + 1] = a[i] * b[i] + a[i + blockDim.x] * b[i + blockDim.x];
sdata[3 * tid + 2] = a[i] * b[i] + a[i + blockDim.x] * b[i + blockDim.x];
__syncthreads();
for (unsigned int s = blockDim.x / 2; s > 96; s >>= 1) {
if (tid < s) {
sdata[3 * tid] += sdata[3 * tid + s];
sdata[3 * tid + 1] += sdata[3 * tid + s + 1];
sdata[3 * tid + 2] += sdata[3 * tid + s + 2];
}
}
if (tid < 32) {
sdata[3 * tid] += sdata[3 * tid + 96];
sdata[3 * tid + 1] += sdata[3 * tid + 97];
sdata[3 * tid + 2] += sdata[3 * tid + 98];
sdata[3 * tid] += sdata[3 * tid + 48];
sdata[3 * tid + 1] += sdata[3 * tid + 49];
sdata[3 * tid + 2] += sdata[3 * tid + 50];
sdata[3 * tid] += sdata[3 * tid + 24];
sdata[3 * tid + 1] += sdata[3 * tid + 25];
sdata[3 * tid + 2] += sdata[3 * tid + 26];
sdata[3 * tid] += sdata[3 * tid + 12];
sdata[3 * tid + 1] += sdata[3 * tid + 13];
sdata[3 * tid + 2] += sdata[3 * tid + 14];
sdata[3 * tid] += sdata[3 * tid + 6];
sdata[3 * tid + 1] += sdata[3 * tid + 7];
sdata[3 * tid + 2] += sdata[3 * tid + 8];
sdata[3 * tid] += sdata[3 * tid + 3];
sdata[3 * tid + 1] += sdata[3 * tid + 4];
sdata[3 * tid + 2] += sdata[3 * tid + 5];
}
if (tid == 0) {
outN[blockIdx.x] = sdata[0];
outD1[blockIdx.x] = sdata[1];
outD2[blockIdx.x] = sdata[2];
}
i += stride;
}
//if (blockSize >= 512) {
//	if (tid < 256) {
//		sndata[tid] += sndata[tid + 256];
//		sd1data[tid] += sd1data[tid + 256];
//		sd2data[tid] += sd2data[tid + 256];
//	} __syncthreads();
//}
//if (blockSize >= 256) {
//	if (tid < 128) {
//		sndata[tid] += sndata[tid + 128];
//		sd1data[tid] += sd1data[tid + 128];
//		sd2data[tid] += sd2data[tid + 128];
//	} __syncthreads();
//}
//if (blockSize >= 128) {
//	if (tid < 64) {
//		sndata[tid] += sndata[tid + 64];
//		sd1data[tid] += sd1data[tid + 64];
//		sd2data[tid] += sd2data[tid + 64];
//	} __syncthreads();
//}
//if (tid < 32) {
//	if (blockSize >= 64) {
//		sndata[tid] += sndata[tid + 32];
//		sd1data[tid] += sd1data[tid + 32];
//		sd2data[tid] += sd2data[tid + 32];
//	}
//	if (blockSize >= 32) {
//		sndata[tid] += sndata[tid + 16];
//		sd1data[tid] += sd1data[tid + 16];
//		sd2data[tid] += sd2data[tid + 16];
//	}
//	if (blockSize >= 16) {
//		sndata[tid] += sndata[tid + 8];
//		sd1data[tid] += sd1data[tid + 8];
//		sd2data[tid] += sd2data[tid + 8];
//	}
//	if (blockSize >= 8) {
//		sndata[tid] += sndata[tid + 4];
//		sd1data[tid] += sd1data[tid + 4];
//		sd2data[tid] += sd2data[tid + 4];
//	}
//	if (blockSize >= 4) {
//		sndata[tid] += sndata[tid + 2];
//		sd1data[tid] += sd1data[tid + 2];
//		sd2data[tid] += sd2data[tid + 2];
//	}
//	if (blockSize >= 2) {
//		sndata[tid] += sndata[tid + 1];
//		sd1data[tid] += sd1data[tid + 1];
//		sd2data[tid] += sd2data[tid + 1];
//	}
//}
}