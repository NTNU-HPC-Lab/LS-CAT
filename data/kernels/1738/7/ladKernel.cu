#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ladKernel(float *a, float *b, float *out, int size) {
extern __shared__ float sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
int stride = blockDim.x * 2 * gridDim.x;
sdata[tid] = 0;
while (i < size) {
sdata[tid] += abs(a[i] - b[i]) + abs(a[i + blockDim.x] - b[i + blockDim.x]);
i += stride;
__syncthreads();
}
for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
if (tid<s)
sdata[tid] += sdata[tid + s];
__syncthreads();

}
if (tid < 32) {
sdata[tid] += sdata[tid + 32];
__syncthreads();
sdata[tid] += sdata[tid + 16];
__syncthreads();
sdata[tid] += sdata[tid + 8];
__syncthreads();
sdata[tid] += sdata[tid + 4];
__syncthreads();
sdata[tid] += sdata[tid + 2];
__syncthreads();
sdata[tid] += sdata[tid + 1];
__syncthreads();
}
if (tid == 0) {
out[blockIdx.x] = sdata[0];
}
}