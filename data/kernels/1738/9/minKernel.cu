#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minKernel(float *array, int size, float* min)
{
extern __shared__ float sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
int stride = blockDim.x * 2 * gridDim.x;
//sdata[tid] = 0;
while (i < size)
{
sdata[tid] = fminf(array[i], array[i + blockDim.x]);
i += stride;
__syncthreads();

}

for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
if (tid < s)
sdata[tid] = fminf(sdata[tid], sdata[tid + s]);
__syncthreads();

}


if (tid < 32) {
sdata[tid] = fminf(sdata[tid], sdata[tid + 32]);
__syncthreads();
sdata[tid] = fminf(sdata[tid], sdata[tid + 16]);
__syncthreads();
sdata[tid] = fminf(sdata[tid], sdata[tid + 8]);
__syncthreads();
sdata[tid] = fminf(sdata[tid], sdata[tid + 4]);
__syncthreads();
sdata[tid] = fminf(sdata[tid], sdata[tid + 2]);
__syncthreads();
sdata[tid] = fminf(sdata[tid], sdata[tid + 1]);
__syncthreads();

}
if (tid == 0) {
min[blockIdx.x] = sdata[0];
}
}