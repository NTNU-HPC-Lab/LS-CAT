#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxKernel(float *array, int size, float* max)
{
extern __shared__ float sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
int stride = blockDim.x * 2 * gridDim.x;
sdata[tid] = 0;
while (i < size)
{
sdata[tid] = fmaxf(array[i], array[i + blockDim.x]);
i += stride;
__syncthreads();

}

for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
if (tid < s)
sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
__syncthreads();

}


if (tid < 32) {
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 32]);
__syncthreads();
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 16]);
__syncthreads();
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 8]);
__syncthreads();
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 4]);
__syncthreads();
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 2]);
__syncthreads();
sdata[tid] = fmaxf(sdata[tid], sdata[tid + 1]);
__syncthreads();

}
if (tid == 0) {
max[blockIdx.x] = sdata[0];
}
}