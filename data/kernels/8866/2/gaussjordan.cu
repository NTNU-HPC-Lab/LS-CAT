#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaussjordan(double *A, double *I, int n, int i) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if (x < n && y < n) {
if (x != i) {
I[x * n + y] -= I[i * n + y] * A[x * n + i];
if (y != i) {
A[x * n + y] -= A[i * n + y] * A[x * n + i];
}
}
}
}