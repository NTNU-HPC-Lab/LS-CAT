#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updZ(float *z, float *f, float tz, float beta, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float a, b, t;

if (px<nx && py<ny)
{
// compute the gradient
a = 0;
b = 0;
if (px<(nx - 1)) a = f[idx + 1] - f[idx];
if (py<(ny - 1)) b = f[idx + nx] - f[idx];

// update z

a = z[2 * idx + 0] + tz*a;
b = z[2 * idx + 1] + tz*b;

t = sqrtf(beta + a*a + b*b);
t = t<1. ? 1. : 1. / t;

z[2 * idx + 0] = a*t;
z[2 * idx + 1] = b*t;
}
}