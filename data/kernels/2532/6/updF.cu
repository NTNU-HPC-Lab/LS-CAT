#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updF(float *f, float *z, float *g, float tf, float lambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float DIVZ;

if (px<nx && py<ny)
{
// compute the divergence
DIVZ = 0;
if ((px<(nx - 1))) DIVZ += z[2 * (idx)+0];
if ((px>0))      DIVZ -= z[2 * (idx - 1) + 0];

if ((py<(ny - 1))) DIVZ += z[2 * (idx)+1];
if ((py>0))      DIVZ -= z[2 * (idx - nx) + 1];

// update f
//f[idx] = (1.-tf*lambda)*f[idx] + tf * DIVZ + tf*lambda*g[idx];
f[idx] = (f[idx] + tf * DIVZ + tf*lambda*g[idx]) / (1 + tf*lambda);
}

}