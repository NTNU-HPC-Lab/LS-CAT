#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate(float *z, float *z0, float tau, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;
if (x<nx && y<ny)
{
float a = z[2 * idx + 0];
float b = z[2 * idx + 1];
float t = 1 / (1 + tau*sqrtf(a*a + b*b));
z[2 * idx + 0] = (z0[2 * idx + 0] + tau*z[2 * idx + 0])*t;
z[2 * idx + 1] = (z0[2 * idx + 1] + tau*z[2 * idx + 1])*t;
}
}