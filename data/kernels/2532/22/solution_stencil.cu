#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solution_stencil(float *zx, float * zy, float *g, float lambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;

float DIVZ;

if (px<nx && py<ny)
{
// compute the divergence
DIVZ = 0;
if ((px<(nx - 1))) DIVZ += zx[(idx)];
if ((px>0))      DIVZ -= zx[(idx - 1)];

if ((py<(ny - 1))) DIVZ += zy[(idx)];
if ((py>0))      DIVZ -= zy[(idx - nx)];

// update f
g[idx] = -DIVZ*lambda + g[idx];
}
}