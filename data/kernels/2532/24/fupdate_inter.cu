#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fupdate_inter(float *z, float *g, float invlambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float DIVZ;

if (px<nx && py<ny)
{
// compute the divergence
DIVZ = 0;
if ((px<(nx - 1))) DIVZ += z[3 * (idx)+0];
if ((px>0))      DIVZ -= z[3 * (idx - 1) + 0];

if ((py<(ny - 1))) DIVZ += z[3 * (idx)+1];
if ((py>0))      DIVZ -= z[3 * (idx - nx) + 1];

// update f
z[3 * idx + 2] = DIVZ - g[idx] * invlambda;
}
}