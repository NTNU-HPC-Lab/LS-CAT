#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solution2(float *f, float lambda, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;


if (x<nx && y<ny)   f[idx] = -f[idx] * lambda;
}