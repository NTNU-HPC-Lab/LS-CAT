#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solution_inter(float *z, float *g, float lambda, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;

if (x<nx && y<ny)   g[idx] = -z[3 * idx + 2] * lambda;
}