#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divergenceL(float *v, float *d, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;

/*
float AX = 0;
if ((idx < N) && (px<(nx-1))) AX += v[2*(idx   )+0];
if ((idx < N) && (px>0))      AX -= v[2*(idx-1 )+0];

if ((idx < N) && (py<(ny-1))) AX += v[2*(idx   )+1];
if ((idx < N) && (py>0))      AX -= v[2*(idx-nx)+1];

if (idx < N)              d[idx] = AX;
*/

if(px<nx && py<ny)
{
float AX = 0;
if((px<(nx - 1))) AX += v[2 * (idx)+0];
if((px>0))      AX -= v[2 * (idx - 1) + 0];

if((py<(ny - 1)))
AX += v[2 * (idx)+1];
if((py>0))
AX -= v[2 * (idx - nx) + 1];

d[idx] = AX;
}
}