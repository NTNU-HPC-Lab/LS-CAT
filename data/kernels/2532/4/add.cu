#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *a, float *b, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;
if (x<nx && y<ny)   b[idx] += a[idx] * .125;

}