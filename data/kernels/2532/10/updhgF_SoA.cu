#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updhgF_SoA(float *f, float *z1, float *z2, float *g, float tf, float invlambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float DIVZ;

if (px<nx && py<ny)
{
// compute the divergence
DIVZ = 0;
if ((px<(nx - 1))) DIVZ += z1[idx];
if ((px>0))      DIVZ -= z1[idx - 1];

if ((py<(ny - 1))) DIVZ += z2[idx];
if ((py>0))      DIVZ -= z2[idx - nx];

// update f
f[idx] = (1 - tf) *f[idx] + tf * (g[idx] + invlambda*DIVZ);
}

}