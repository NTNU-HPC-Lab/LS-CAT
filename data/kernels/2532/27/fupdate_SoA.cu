#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fupdate_SoA(float *f, float *z1, float *z2, float *g, float invlambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float DIVZ;

if (px<nx && py<ny)
{
// compute the divergence
DIVZ = 0;
float Z1c = z1[(idx)];
float Z2c = z2[(idx)];
float Z1l = z1[(idx - 1)];
float Z2d = z2[(idx - nx)];
if (!(px == (nx - 1))) DIVZ += Z1c;
if (!(px == 0))      DIVZ -= Z1l;
if (!(py == (ny - 1))) DIVZ += Z2c;
if (!(py == 0))      DIVZ -= Z2d;

// update f
f[idx] = DIVZ - g[idx] * invlambda;
}
}