#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lap(float *a, float *b, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;

if (x<nx && y<ny)
{
float AX = 0, BX = 0;
if (x>0)   { BX += a[idx - 1]; AX++; }
if (y>0)   { BX += a[idx - nx]; AX++; }
if (x<nx - 1){ BX += a[idx + 1]; AX++; }
if (y<ny - 1){ BX += a[idx + nx]; AX++; }
b[idx] = -AX*a[idx] + BX;
}
}