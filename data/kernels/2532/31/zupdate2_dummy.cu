#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate2_dummy(float *z1, float *z2, float *f, float tau, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float a, b, t;

if (px<nx && py<ny)
{
// compute the gradient
a = 0;
b = 0;
float fc = f[idx];
//		float fr=f[idx+1];
//		float fu=f[idx+nx];
//    if (!(px==(nx-1))) a = fr - fc;
//     if (!(py==(ny-1))) b = fu - fc;
a = fc;
b = fc;

// update z
t = 1 / (1 + tau*sqrtf(a*a + b*b));
z1[idx] = (z1[idx] + tau*a)*t;
z2[idx] = (z2[idx] + tau*b)*t;
}
}