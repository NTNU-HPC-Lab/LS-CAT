#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate2(float *z, float *f, float tau, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float a, b, t;

if (px<nx && py<ny)
{
// compute the gradient
a = 0;
b = 0;
float fc = f[idx];
if (!(px == (nx - 1))) a = f[idx + 1] - fc;
if (!(py == (ny - 1))) b = f[idx + nx] - fc;

// update z
t = 1 / (1 + tau*sqrtf(a*a + b*b));
z[2 * idx + 0] = (z[2 * idx + 0] + tau*a)*t;
z[2 * idx + 1] = (z[2 * idx + 1] + tau*b)*t;
}
}