#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub0(float *div0, float *div, float *g, float lambda, int nx, int ny)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int idx = x + y*nx;
if (x<nx && y<ny)   div[idx] = div0[idx] - g[idx] / lambda;
}