#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate_inter(float *z, float tau, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
float a, b, t;

if (px<nx && py<ny)
{
// compute the gradient
a = 0;
b = 0;
if (px<(nx - 1)) a = z[3 * (idx + 1) + 2] - z[3 * idx + 2];
if (py<(ny - 1)) b = z[3 * (idx + nx) + 2] - z[3 * idx + 2];

// update z
t = 1 / (1 + tau*sqrtf(a*a + b*b));
z[3 * idx + 0] = (z[3 * idx + 0] + tau*a)*t;
z[3 * idx + 1] = (z[3 * idx + 1] + tau*b)*t;
}
}