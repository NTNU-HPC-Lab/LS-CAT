#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient(float *u, float *g, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
/*
if  (idx<N)
{
g[2*idx+0] = 0;
g[2*idx+1] = 0;
}
if ((idx< N) && px<(nx-1)) g[2*idx+0] = u[idx+1 ] - u[idx];
if ((idx< N) && py<(ny-1)) g[2*idx+1] = u[idx+nx] - u[idx];
*/
if (px<nx && py<ny)
{
g[2 * idx + 0] = 0;
g[2 * idx + 1] = 0;
if (px<(nx - 1)) g[2 * idx + 0] = u[idx + 1] - u[idx];
if (py<(ny - 1)) g[2 * idx + 1] = u[idx + nx] - u[idx];
}
//a[idx] =0;
}