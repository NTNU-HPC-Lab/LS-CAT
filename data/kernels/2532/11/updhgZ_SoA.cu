#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updhgZ_SoA(float *z1, float *z2, float *f, float tz, float lambda, int nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;

if (px<nx && py<ny)
{
// compute the gradient
float a = 0;
float b = 0;
float fc = f[idx];
if (px<(nx - 1)) a = f[idx + 1] - fc;
if (py<(ny - 1)) b = f[idx + nx] - fc;

// update z

a = z1[idx] + tz*lambda*a;
b = z2[idx] + tz*lambda*b;

// project
float t = 0;
t = sqrtf(a*a + b*b);
t = (t <= 1 ? 1. : t);

z1[idx] = a / t;
z2[idx] = b / t;
}
}