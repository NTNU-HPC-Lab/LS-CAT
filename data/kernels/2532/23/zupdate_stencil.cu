#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate_stencil(float *zx, float *zy, float *zoutx, float *zouty, float *g, float tau, float invlambda, int  nx, int ny)
{
int px = blockIdx.x * blockDim.x + threadIdx.x;
int py = blockIdx.y * blockDim.y + threadIdx.y;
int idx = px + py*nx;
int tidx, tpx, tpy;
float a, b, t;
float DIVZ;

/* compute simultaneously
f= div z -g /lambda at the positions
right, center north*/
float fr = 0, fc = 0, fu = 0;

////////////////////////////////////////////////////////
//
//		(zul)		(zu)
//
//					____
//		(zl)		|zc|		(zr)
//					----
//
//					(zd)		(zdr)
//
//		if the pixel is not inside the region then put 0
//
//		fc = z1c - z1l + z2c - z2d
//		fr = z1r - z1c + z2r - z2dr
//		fu = z1u - z1ul + z2u - z2c
//
////////////////////////////////////////////////////////

tidx = idx;
tpx = px;
tpy = py;
if (tpx<nx && tpy<ny)
{
// compute the divergence
DIVZ = 0;
if ((tpx<(nx - 1))) DIVZ += zx[tidx];
if ((tpx>0))      DIVZ -= zx[tidx - 1];

if ((tpy<(ny - 1))) DIVZ += zy[tidx];
if ((tpy>0))      DIVZ -= zy[tidx - nx];

fc = DIVZ;
}
////////////////////////////////////////////////////////

tidx = idx + 1;
tpx = px + 1;
tpy = py;
if (tpx<nx && tpy<ny)
{
// compute the divergence
DIVZ = 0;
if ((tpx<(nx - 1))) DIVZ += zx[tidx];
if ((tpx>0))      DIVZ -= zx[tidx - 1];

if ((tpy<(ny - 1))) DIVZ += zy[tidx];
if ((tpy>0))      DIVZ -= zy[tidx - nx];

fr = DIVZ;
}
////////////////////////////////////////////////////////

tidx = idx + nx;
tpx = px;
tpy = py + 1;
if (tpx<nx && tpy<ny)
{
// compute the divergence
DIVZ = 0;
if ((tpx<(nx - 1))) DIVZ += zx[tidx];
if ((tpx>0))      DIVZ -= zx[tidx - 1];

if ((tpy<(ny - 1))) DIVZ += zy[tidx];
if ((tpy>0))      DIVZ -= zy[tidx - nx];

fu = DIVZ;
}

fr = fr - g[idx + 1] * invlambda;
fc = fc - g[idx] * invlambda;
fu = fu - g[idx + nx] * invlambda;

////////////////////////////////////////////////////////

if (px<nx && py<ny)
{
// compute the gradient
a = 0;
b = 0;
if (px<(nx - 1)) a = fr - fc;
if (py<(ny - 1)) b = fu - fc;

// update z
t = 1 / (1 + tau*sqrtf(a*a + b*b));
zoutx[idx] = (zx[idx] + tau*a)*t;
zouty[idx] = (zy[idx] + tau*b)*t;
}
}