#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
__global__ void computeDistances(int numInstances, int numAttributes, float* dataset, float* distances)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;
int row = tid / numInstances; // instance1Index
int column = tid - ((tid / numInstances) * numInstances); //instance2Index
if ((tid < numInstances * numInstances))
{
float sum = 0;
int instance1 = row * numAttributes;
int instance2 = column * numAttributes;
for (int atIdx = 1; atIdx < numAttributes; atIdx++) // start at 1 so we don't compare the id of each city
{
sum += ((dataset[instance1 + atIdx] - dataset[instance2 + atIdx]) * (dataset[instance1 + atIdx] - dataset[instance2 + atIdx]));
}
distances[row * numInstances + column] = (float) sqrt(sum);
distances[column * numInstances + row] = distances[row * numInstances + column]; //set the distance for the other half of the pair we just computed
}
}