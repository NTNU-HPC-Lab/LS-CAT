#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_addcdiv(float *data, float value, float *src1, float *src2, long size)
{
long k = (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x) + threadIdx.x;

if(k < size)
data[k] += value*src1[k]/src2[k];
}