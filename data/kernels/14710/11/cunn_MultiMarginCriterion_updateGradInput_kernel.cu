#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MultiMarginCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage)
{
__shared__ float buffer[MULTIMARGIN_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *gradInput_k = gradInput + k*dim;
int target_k = ((int)target[k])-1;
float input_target_k = input_k[target_k];
float g = (sizeaverage ? 1./((float)dim) : 1.);

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float z = 1 - input_target_k + input_k[i];
if(i == target_k)
continue;

if(z > 0)
{
buffer[threadIdx.x] -= g;
gradInput_k[i] = g;
}
else
gradInput_k[i] = 0;
}

__syncthreads();

// reduce
if (threadIdx.x == 0)
{
float gradInput_target_k = 0;
for (int i=0; i<blockDim.x; i++)
gradInput_target_k += buffer[i];
gradInput_k[target_k] = gradInput_target_k;
}
}