#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize, float* gpuFilter, int fsx, int fsy, int fsz, float* gpuBuffer, int ringBufferIdx, int bsx, int bsy, int bsz, int fs_xy, int fn)
{

// Calculate filter idx
int filterPos = threadIdx.x + blockIdx.x * blockDim.x;

// Per block shared memory
__shared__ uint8_t gpuEventListSharedX[MAX_SHARED_GPU_EVENTS];
__shared__ uint8_t gpuEventListSharedY[MAX_SHARED_GPU_EVENTS];
// How many runs do we need to process all events
int processingRuns = ceilf((float)gpuEventListSize/MAX_SHARED_GPU_EVENTS);
// Events for each thread to read
int eventReadsPerThread = ceilf((float)MAX_SHARED_GPU_EVENTS/blockDim.x);
// Offset n global event buffer
int globalEventIdx = threadIdx.x;

// Idx valid
if (filterPos < fn) {
// Read filter coefficient from global memory
float filterVal = gpuFilter[filterPos];
// Compute x,y,z coodinates in buffer
int fz = filterPos / fs_xy;
int fxy = filterPos % fs_xy;
int fy = fxy / fsx;
int fx = fxy % fsx;

// Convert buffer coordinates (mirror all axes -> convolution instead of correlation)
// Origin for mirroring is x = w/2, y = h/2, z = 0
int bz = ((ringBufferIdx + (fsz - 1) - fz ) % bsz);
int bx_tmp = fsx / 2 - fx;
int by_tmp = fsy / 2 - fy;
int bPos_tmp = bz*bsy*bsx;

int sharedEventCnt = MAX_SHARED_GPU_EVENTS;
// Iterate over event list in blocks, stored in shared memory
for(int runIdx = 0; runIdx<processingRuns; runIdx++) {
// Last run ? Compute size of shared event list
if(runIdx+1 == processingRuns) {
sharedEventCnt = gpuEventListSize % MAX_SHARED_GPU_EVENTS;
}
// Compute index in shared memory
int localEventIdx = threadIdx.x;

// Fill the shared memory either with MAX_SHARED_GPU_EVENTS
// or use each thread mutlible times
for(int i = 0; i < eventReadsPerThread; i++) {
// Valid indices
if(localEventIdx >= sharedEventCnt)
break;
// Load event into shared memory by using one thread per event
gpuEventListSharedX[localEventIdx] = gpuEventsX[globalEventIdx];
gpuEventListSharedY[localEventIdx] = gpuEventsY[globalEventIdx];

// Goto next event for which this thread is responsible
localEventIdx += blockDim.x;
globalEventIdx += blockDim.x;
}

// Synchronize threads and wait until shared memory is filled
// TODO: Deadlock possible?
// At least one thread in each warp should hit that barrier to continue!
// Bad relationship between shared event list size and block size could cause problems ?!
__syncthreads();

// Iterate over every event block in shared memory
for(localEventIdx = 0; localEventIdx < sharedEventCnt; localEventIdx++) {
// Compute corresponding buffer coordinate
int bx = bx_tmp + gpuEventListSharedX[localEventIdx];
int by = by_tmp + gpuEventListSharedY[localEventIdx];

// Check for valid buffer position (filp buffer z)
if(bx >= 0 && bx < bsx && by >= 0 && by < bsy) {
int bufferPos = bPos_tmp + by*bsx + bx;
// Add each filter coefficient to the global buffer
atomicAdd(gpuBuffer + bufferPos,filterVal);
}
}
}
}
}