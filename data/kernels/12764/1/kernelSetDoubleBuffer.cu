#include "hip/hip_runtime.h"
#include "includes.h"

/**
* @brief cudaCreateBuffer Allocates a cuda buffer and stops the programm on error.
* @param size
* @return
*/
__global__ void kernelSetDoubleBuffer(float* gpuBuffPtr, float v, size_t size)
{
int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index < size)
gpuBuffPtr[index] = v;
}