#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelReadMotionEnergyAsync(float* gpuConvBufferl1, float* gpuConvBufferl2, int ringBufferIdx, int bsx, int bsy, int n, float* gpuEnergyBuffer)
{
int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
if(bufferPos < n) {
// Offset in ringbuffer
int bufferPosConv = bufferPos + ringBufferIdx*bsx*bsy;
// Get answer from two corresponding buffers and compute motion energy
float l1 = gpuConvBufferl1[bufferPosConv];
float l2 = gpuConvBufferl2[bufferPosConv];

// Compute motion energy
gpuEnergyBuffer[bufferPos] = sqrt(l1*l1+l2*l2);
}
}