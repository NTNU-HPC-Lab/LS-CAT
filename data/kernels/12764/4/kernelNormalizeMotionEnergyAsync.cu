#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelNormalizeMotionEnergyAsync(int bsx, int bsy, int n, float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm, float* gpuEnergyBuffer)
{
int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
float sigmaNorm2_2 = 2*sigmaNorm*sigmaNorm;
if(bufferPos < n) {
int bx,by;
int bxy = bufferPos / (bsx*bsy);
bx = bxy % bsx;
by = bxy / bsx;
// Read energy
float I = gpuEnergyBuffer[bufferPos];
float q_i = 0;
// Normalize over 5x5 region
for(int y = -2; y <= 2; y++) {
int by_ = by + y;

if(by_ < 0 || by_ >= bsy)
continue;

for(int x = -2; x <= 2; x++) {
int bx_ = bx + x;

if(bx_ < 0 || bx_ >= bsx ||
(bx == bx_ && by == by_))
continue;
// TODO
// Each thread computes the same
float gaus = 1/(sigmaNorm2_2*M_PI)* exp(-(bx_*bx_ + by_*by_)/sigmaNorm2_2);
// TODO Use shared memory to avoid extra global memory access
q_i += gpuEnergyBuffer[by_*bsx+bx_]*gaus;
}
}
q_i /= alphaQNorm;

// Compute p_i
float p_i = (I*betaNorm)/(alphaPNorm + I + q_i);

// Use normalized value
gpuEnergyBuffer[bufferPos] = p_i;
}
}