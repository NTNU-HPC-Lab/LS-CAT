#include "hip/hip_runtime.h"
#include "includes.h"
/*
* test.cpp
* (c) 2015
* Author: Jim Fan
* See below link for how to support C++11 in eclipse
* http://scrupulousabstractions.tumblr.com/post/36441490955/eclipse-mingw-builds
*/

#ifdef is_CUDA
#endif

__global__ void testkernel()
{
double p = threadIdx.x + 66;
for (int i = 0; i < 30000000; ++i)
p += i / p - std::sqrt(p);

printf("thread %d; block %d\n", threadIdx.x, blockIdx.x);
}