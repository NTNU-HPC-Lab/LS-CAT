#include "hip/hip_runtime.h"
#include "includes.h"





using namespace std;

#define MEDIAN_DIMENSION  3 // For matrix of 3 x 3. We can Use 5 x 5 , 7 x 7 , 9 x 9......
#define MEDIAN_LENGTH 9   // Shoul be  MEDIAN_DIMENSION x MEDIAN_DIMENSION = 3 x 3

#define BLOCK_WIDTH 16  // Should be 8 If matrix is of larger then of 5 x 5 elese error occur as " uses too much shared data "  at surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH]
#define BLOCK_HEIGHT 16// Should be 8 If matrix is of larger then of 5 x 5 elese error occur as " uses too much shared data "  at surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH]


__global__ void MedianFilter_gpu(unsigned short *Device_ImageData, int Image_Width, int Image_Height) {

__shared__ unsigned short surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH];

int iterator;
const int Half_Of_MEDIAN_LENGTH = (MEDIAN_LENGTH / 2) + 1;
int StartPoint = MEDIAN_DIMENSION / 2;
int EndPoint = StartPoint + 1;

const int x = blockDim.x * blockIdx.x + threadIdx.x;
const int y = blockDim.y * blockIdx.y + threadIdx.y;

const int tid = threadIdx.y*blockDim.y + threadIdx.x;

if (x >= Image_Width || y >= Image_Height)
return;

//Fill surround with pixel value of Image in Matrix Pettern of MEDIAN_DIMENSION x MEDIAN_DIMENSION
if (x == 0 || x == Image_Width - StartPoint || y == 0
|| y == Image_Height - StartPoint) {
}
else {
iterator = 0;
for (int r = x - StartPoint; r < x + (EndPoint); r++) {
for (int c = y - StartPoint; c < y + (EndPoint); c++) {
surround[tid][iterator] = *(Device_ImageData + (c*Image_Width) + r);
iterator++;
}
}
//Sort the Surround Array to Find Median. Use Bubble Short  if Matrix oF 3 x 3 Matrix
//You can use Insertion commented below to Short Bigger Dimension Matrix

////      bubble short //

for (int i = 0; i<Half_Of_MEDIAN_LENGTH; ++i)
{
// Find position of minimum element
int min = i;
for (int l = i + 1; l<MEDIAN_LENGTH; ++l)
if (surround[tid][l] <surround[tid][min])
min = l;
// Put found minimum element in its place
unsigned short  temp = surround[tid][i];
surround[tid][i] = surround[tid][min];
surround[tid][min] = temp;
}//bubble short  end

//////insertion sort start   //

/*int t,j,i;
for ( i = 1 ; i< MEDIAN_LENGTH ; i++) {
j = i;
while ( j > 0 && surround[tid][j] < surround[tid][j-1]) {
t= surround[tid][j];
surround[tid][j]= surround[tid][j-1];
surround[tid][j-1] = t;
j--;
}
}*/

////insertion sort end



*(Device_ImageData + (y*Image_Width) + x) = surround[tid][Half_Of_MEDIAN_LENGTH - 1];   // it will give value of surround[tid][4] as Median Value if use 3 x 3 matrix
__syncthreads();
}
}