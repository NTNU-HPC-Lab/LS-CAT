#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *g_idata, int searchedNumber, int *ok) {

int i = blockIdx.x * blockDim.x + threadIdx.x;
//printf("%d ", i);

__syncthreads();
//printf("%d %d///", g_idata[i], searchedNumber);
if (g_idata[i] == searchedNumber) {
printf("Found %d on %d position %d", searchedNumber, i, *ok);
*ok = i;
}
}