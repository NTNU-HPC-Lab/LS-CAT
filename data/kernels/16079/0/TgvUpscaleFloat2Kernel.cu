#include "hip/hip_runtime.h"
#include "includes.h"

/// scalar field to upscale
texture<float, hipTextureType2D, hipReadModeElementType> texCoarse;
texture<float2, hipTextureType2D, hipReadModeElementType> texCoarseFloat2;

__global__
__global__ void TgvUpscaleFloat2Kernel(int width, int height, int stride, float scale, float2 *out)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix >= width || iy >= height) return;

float x = ((float)ix + 0.5f) / (float)width;
float y = ((float)iy + 0.5f) / (float)height;

// exploit hardware interpolation
// and scale interpolated vector to match next pyramid level resolution
float2 src = tex2D(texCoarseFloat2, x, y);
out[ix + iy * stride].x = src.x * scale;
out[ix + iy * stride].y = src.y * scale;
}