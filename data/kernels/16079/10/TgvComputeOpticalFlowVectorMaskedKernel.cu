#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvComputeOpticalFlowVectorMaskedKernel(const float *u, const float2 *tv2, float* mask, int width, int height, int stride, float2 *warpUV)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
if (mask[pos] == 0.0f) return;

float us = u[pos];
float2 tv2s = tv2[pos];
warpUV[pos].x = us * tv2s.x;
warpUV[pos].y = us * tv2s.y;
}