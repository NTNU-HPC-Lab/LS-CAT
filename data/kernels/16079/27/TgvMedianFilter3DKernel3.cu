#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvMedianFilter3DKernel3(float* X, float* Y, float *Z, int width, int height, int stride, float *X1, float *Y1, float *Z1)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float mX[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
float mY[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
float mZ[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

for (int j = 0; j < 3; j++) {
for (int i = 0; i < 3; i++) {
//get values
int col = (ix + i - 1);
int row = (iy + j - 1);
int index = j * 3 + i;
if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
mX[index] = X[col + stride * row];
mY[index] = Y[col + stride * row];
mZ[index] = Z[col + stride * row];
}
else if ((col < 0) && (row >= 0) && (row < height)) {
mX[index] = X[stride*row];
mY[index] = Y[stride*row];
mZ[index] = Z[stride*row];
}
else if ((col > width) && (row >= 0) && (row < height)) {
mX[index] = X[width - 1 + stride * row];
mY[index] = Y[width - 1 + stride * row];
mZ[index] = Z[width - 1 + stride * row];
}
else if ((col >= 0) && (col < width) && (row < 0)) {
mX[index] = X[col];
mY[index] = Y[col];
mZ[index] = Z[col];
}
else if ((col >= 0) && (col < width) && (row > height)) {
mX[index] = X[col + stride * (height - 1)];
mY[index] = Y[col + stride * (height - 1)];
mZ[index] = Z[col + stride * (height - 1)];
}
//solve gaussian
}
}

float tmpX, tmpY, tmpZ;
for (int j = 0; j < 5; j++) {
for (int i = j + 1; i < 9; i++) {
if (mX[j] > mX[i]) {
//Swap the variables.
tmpX = mX[j];
mX[j] = mX[i];
mX[i] = tmpX;
}
if (mY[j] > mY[i]) {
//Swap the variables.
tmpY = mY[j];
mY[j] = mY[i];
mY[i] = tmpY;
}
if (mZ[j] > mZ[i]) {
//Swap the variables.
tmpZ = mZ[j];
mZ[j] = mZ[i];
mZ[i] = tmpZ;
}
}
}

X1[pos] = mX[4];
Y1[pos] = mY[4];
Z1[pos] = mZ[4];
}