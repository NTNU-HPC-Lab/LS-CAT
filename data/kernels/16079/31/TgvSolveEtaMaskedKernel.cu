#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvSolveEtaMaskedKernel(float* mask, float alpha0, float alpha1, float* atensor, float *btensor, float* ctensor, float* etau, float* etav1, float* etav2, int width, int height, int stride)
{
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
if (mask[pos] == 0.0f) return;

float a = atensor[pos];
float b = btensor[pos];
float c = ctensor[pos];

etau[pos] = (a*a + b * b + 2 * c*c + (a + c)*(a + c) + (b + c)*(b + c)) * (alpha1 * alpha1);
etav1[pos] = (alpha1 * alpha1)*(b * b + c * c) + 4 * alpha0 * alpha0;
etav2[pos] = (alpha1 * alpha1)*(a * a + c * c) + 4 * alpha0 * alpha0;
}