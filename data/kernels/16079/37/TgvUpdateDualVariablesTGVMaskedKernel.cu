#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvUpdateDualVariablesTGVMaskedKernel(float* mask, float* u_, float2 *v_, float alpha0, float alpha1, float sigma, float eta_p, float eta_q, float* a, float* b, float*c, float4* grad_v, float2* p, float4* q, int width, int height, int stride)
{
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
if (mask[pos] == 0.0f) return;

//p[pos] = make_float2(0.0f, 0.0f);
//q[pos] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

int right = (ix + 1) + iy * stride;
int down = ix + (iy + 1) * stride;
int left = (ix - 1) + iy * stride;
int up = ix + (iy - 1) * stride;
float maskRight, maskDown;
if (ix + 1 >= width) {
maskRight = 0.0f;
}
else maskRight = mask[right];
if (iy + 1 >= height) {
maskDown = 0.0f;
}
else maskDown = mask[down];

float u_pos = u_[pos];
float2 v_pos = v_[pos];

//u_x = dxp(u_) - v_(:, : , 1);
float u_x, u_y;
if (maskRight != 0.0f)
u_x = u_[right] - u_pos - v_pos.x;
else
u_x = u_pos - u_[left] - v_pos.x;
//u_y = dyp(u_) - v_(:, : , 2);
if (maskDown != 0.0f)
u_y = u_[down] - u_pos - v_pos.y;
else
u_y = u_pos - u_[up] - v_pos.y;

//du_tensor_x = a.*u_x + c.*u_y;
float du_tensor_x = a[pos] * u_x + c[pos] * u_y;
//du_tensor_y = c.*u_x + b.*u_y;
float du_tensor_y = c[pos] * u_x + b[pos] * u_y;

float2 ppos;
//p(:, : , 1) = p(:, : , 1) + alpha1*sigma / eta_p.*du_tensor_x;
ppos.x = p[pos].x + (alpha1*sigma / eta_p) * du_tensor_x;
//p(:, : , 2) = p(:, : , 2) + alpha1*sigma / eta_p.*du_tensor_y;
ppos.y = p[pos].y + (alpha1*sigma / eta_p) * du_tensor_y;

//projection
//reprojection = max(1.0, sqrt(p(:, : , 1). ^ 2 + p(:, : , 2). ^ 2));
float reprojection = sqrtf(ppos.x * ppos.x + ppos.y * ppos.y);
if (reprojection < 1.0f) {
reprojection = 1.0f;
}
//p(:, : , 1) = p(:, : , 1). / reprojection;
p[pos].x = ppos.x / reprojection;
//p(:, : , 2) = p(:, : , 2). / reprojection;
p[pos].y = ppos.y / reprojection;

//grad_v(:, : , 1) = dxp(v_(:, : , 1));
float4 grad_v_pos;
if (maskRight != 0.0f)
grad_v_pos.x = v_[right].x - v_pos.x;
else
grad_v_pos.x = v_pos.x - v_[left].x;

//grad_v(:, : , 2) = dyp(v_(:, : , 2));
if (maskDown != 0.0f)
grad_v_pos.y = v_[down].y - v_pos.y;
else
grad_v_pos.y = v_pos.y - v_[up].y;

//grad_v(:, : , 3) = dyp(v_(:, : , 1));
if (maskDown != 0.0f)
grad_v_pos.z = v_[down].x - v_pos.x;
else
grad_v_pos.z = v_pos.x - v_[up].x;

//grad_v(:, : , 4) = dxp(v_(:, : , 2));
if (maskRight != 0.0f)
grad_v_pos.w = v_[right].y - v_pos.y;
else
grad_v_pos.w = v_pos.y - v_[left].y;

grad_v[pos] = grad_v_pos;

//q = q + alpha0*sigma / eta_q.*grad_v;
float ase = alpha0 * sigma / eta_q;
float4 qpos;
qpos.x = q[pos].x + ase * grad_v_pos.x;
qpos.y = q[pos].y + ase * grad_v_pos.y;
qpos.z = q[pos].z + ase * grad_v_pos.z;
qpos.w = q[pos].w + ase * grad_v_pos.w;

//reproject = max(1.0, sqrt(q(:, : , 1). ^ 2 + q(:, : , 2). ^ 2 + q(:, : , 3). ^ 2 + q(:, : , 4). ^ 2));
float reproject = sqrtf(qpos.x * qpos.x + qpos.y * qpos.y + qpos.z * qpos.z + qpos.w * qpos.w);
if (reproject < 1.0f) {
reproject = 1.0f;
}
//q(:, : , 1) = q(:, : , 1). / reproject;
q[pos].x = qpos.x / reproject;
//q(:, : , 2) = q(:, : , 2). / reproject;
q[pos].y = qpos.y / reproject;
//q(:, : , 3) = q(:, : , 3). / reproject;
q[pos].z = qpos.z / reproject;
//q(:, : , 4) = q(:, : , 4). / reproject;
q[pos].w = qpos.w / reproject;
}