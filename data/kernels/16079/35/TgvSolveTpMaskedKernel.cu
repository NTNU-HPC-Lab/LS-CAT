#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvSolveTpMaskedKernel(float* mask, float*a, float *b, float*c, float2* p, float2* Tp, int width, int height, int stride) {
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
if (mask[pos] == 0.0f) return;

Tp[pos].x = a[pos] * p[pos].x + c[pos] * p[pos].y;
Tp[pos].y = c[pos] * p[pos].x + b[pos] * p[pos].y;
}