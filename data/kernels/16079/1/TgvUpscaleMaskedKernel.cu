#include "hip/hip_runtime.h"
#include "includes.h"

/// scalar field to upscale
texture<float, hipTextureType2D, hipReadModeElementType> texCoarse;
texture<float2, hipTextureType2D, hipReadModeElementType> texCoarseFloat2;

__global__
__global__ void TgvUpscaleMaskedKernel(float * mask, int width, int height, int stride, float scale, float *out)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
//if (mask[pos] == 0.0f) return;

float x = ((float)ix + 0.5f) / (float)width;
float y = ((float)iy + 0.5f) / (float)height;

// exploit hardware interpolation
// and scale interpolated vector to match next pyramid level resolution
out[pos] = tex2D(texCoarse, x, y) * scale;

//if (ix >= width || iy >= height) return;

//// exploit hardware interpolation
//// and scale interpolated vector to match next pyramid level resolution
//out[ix + iy * stride] = tex2D(texCoarse, x, y) * scale;
}