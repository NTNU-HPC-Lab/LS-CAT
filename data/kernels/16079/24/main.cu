#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "TgvConvertKBKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float2 *disparity = NULL;
hipMalloc(&disparity, XSIZE*YSIZE);
float focalx = 1;
float focaly = 1;
float cx = 1;
float cy = 1;
float d1 = 2;
float d2 = 2;
float d3 = 2;
float d4 = 1;
float t1 = 1;
float t2 = 1;
float t3 = 1;
float3 *X = NULL;
hipMalloc(&X, XSIZE*YSIZE);
float *depth = NULL;
hipMalloc(&depth, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
TgvConvertKBKernel<<<gridBlock,threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
TgvConvertKBKernel<<<gridBlock,threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
TgvConvertKBKernel<<<gridBlock,threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}