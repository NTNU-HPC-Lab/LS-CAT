#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvThresholdingL1MaskedKernel(float2* Tp, float* u_, float* Iu, float* Iz, float* mask, float lambda, float tau, float* eta_u, float* u, float* us, int width, int height, int stride)
{
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy >= height) && (ix >= width)) return;
int pos = ix + iy * stride;
if (mask[pos] == 0.0f) return;

int right = (ix + 1) + iy * stride;
int down = ix + (iy + 1) * stride;
int left = (ix - 1) + iy * stride;
int up = ix + (iy - 1) * stride;

float maskRight, maskLeft, maskUp, maskDown;

if (ix + 1 >= width) maskRight = 0.0f;
else maskRight = mask[right];

if (ix - 1 < 0) maskLeft = 0.0f;
else maskLeft = mask[left];

if (iy + 1 >= height) maskDown = 0.0f;
else maskDown = mask[down];

if (iy - 1 < 0) maskUp = 0.0f;
else maskUp = mask[up];

//div_p = dxm(Tp(:, : , 1)) + dym(Tp(:, : , 2));
float div_p;
float dxmTp, dymTp;

//if ((ix - 1) >= 0)
if ((maskLeft != 0.0f) && (maskRight != 0.0f))
dxmTp = Tp[pos].x - Tp[left].x;
else if (maskRight == 0.0f)
dxmTp = -Tp[left].x;
else
dxmTp = Tp[pos].x;

if ((maskUp != 0.0f) && (maskDown != 0.0f))
dymTp = Tp[pos].y - Tp[up].y;
else if (maskDown == 0.0f)
dymTp = -Tp[up].y;
else
dymTp = Tp[pos].y;

div_p = dxmTp + dymTp;

//tau_eta_u = tau. / eta_u;
float tau_eta_u;
if (eta_u[pos] == 0) {
tau_eta_u = tau;
}
else {
tau_eta_u = tau / eta_u[pos];
}

// Thresholding
float uhat = u_[pos] + tau_eta_u * div_p;

float dun = (uhat - u[pos]);

float Ius = Iu[pos];
float rho = Ius * dun + Iz[pos];

float upper = lambda * tau_eta_u*(Ius*Ius);
float lower = -lambda * tau_eta_u*(Ius*Ius);
float du;

if ((rho <= upper) && (rho >= lower)) {
if (Ius == 0) {
du = dun;
}
else {
du = dun - rho / Ius;
}
}
else if (rho < lower) {
du = dun + lambda * tau_eta_u*Ius;
}
else if (rho > upper) {
du = dun - lambda * tau_eta_u*Ius;
}

us[pos] = u[pos] + du;
}