#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvSolveTpKernel(float*a, float *b, float*c, float2* p, float2* Tp, int width, int height, int stride) {
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy < height) && (ix < width))
{
int pos = ix + iy * stride;

Tp[pos].x = a[pos] * p[pos].x + c[pos] * p[pos].y;
Tp[pos].y = c[pos] * p[pos].x + b[pos] * p[pos].y;
}
}