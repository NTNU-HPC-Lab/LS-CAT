#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvCloneKernel2(float2* dst, float2* src, int width, int height, int stride) {
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy < height) && (ix < width))
{
int pos = ix + iy * stride;
dst[pos] = src[pos];
}
}