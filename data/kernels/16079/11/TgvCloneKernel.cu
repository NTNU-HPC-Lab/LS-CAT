#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvCloneKernel(float* dst, float* src, int width, int height, int stride) {
int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

if ((iy < height) && (ix < width))
{
int pos = ix + iy * stride;
dst[pos] = src[pos];
}
}