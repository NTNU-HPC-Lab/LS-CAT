#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvComputeOpticalFlowVectorKernel(const float *u, const float2 *tv2, int width, int height, int stride, float2 *warpUV)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;

const int pos = ix + iy * stride;

if (ix >= width || iy >= height) return;

float us = u[pos];
float2 tv2s = tv2[pos];
warpUV[pos].x = us * tv2s.x;
warpUV[pos].y = us * tv2s.y;
}