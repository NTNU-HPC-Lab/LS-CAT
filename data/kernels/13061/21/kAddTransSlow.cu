#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned  int i) {
return height * (i % width) + i / width;
}
__global__ void kAddTransSlow(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int numEls, float scaleA, float scaleB) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
//    const unsigned int idx = blockIdx.y * height + blockIdx.x * blockDim.x  + threadIdx.y*blockDim.x + threadIdx.x;
for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = scaleA * a[i] + scaleB * b[getTransArrayIndex(width, height, i)];
}
}