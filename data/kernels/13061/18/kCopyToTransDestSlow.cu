#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned  int i) {
return height * (i % width) + i / width;
}
__global__ void kCopyToTransDestSlow(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcJumpWidth, unsigned int destJumpHeight, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < numElements)
destStart[getTransArrayIndex(srcCopyWidth, destJumpHeight, idx)] = srcStart[(idx / srcCopyWidth) * srcJumpWidth + idx % srcCopyWidth];
}