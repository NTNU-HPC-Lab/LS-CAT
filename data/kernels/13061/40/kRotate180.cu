#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRotate180(float* filters, float* targets, const int filterSize) {
//   __shared__ float shFilter[16][16];

const int filtIdx = blockIdx.x;
const int readStart = MUL24(MUL24(filterSize, filterSize), filtIdx);
filters += readStart;
targets += readStart;

for(int y = threadIdx.y; y < filterSize; y += 16) {
for(int x = threadIdx.x; x < filterSize; x += 16) {
const int writeX = filterSize - 1 - x;
const int writeY = filterSize - 1 - y;

targets[MUL24(writeY, filterSize) + writeX] = filters[MUL24(y, filterSize) + x];
}
}
}