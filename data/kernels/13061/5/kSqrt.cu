#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqrt(float* gData, float* target, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
target[i] = sqrtf(gData[i]);
}