#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivideTransFast(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int bJumpWidth) {
const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int idx = idxY * width + idxX;

__shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

const unsigned int bBlockReadStart = blockDim.x * blockIdx.x * bJumpWidth + blockIdx.y * blockDim.y;

smem[threadIdx.x][threadIdx.y] = b[bBlockReadStart + threadIdx.y * bJumpWidth + threadIdx.x];
__syncthreads();

dest[idx] = __fdividef(a[idx], smem[threadIdx.y][threadIdx.x]);
}