#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCopyToTransDestFast(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcCopyHeight, unsigned int srcJumpSize, unsigned int destJumpSize) {
//    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
//    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;

//    if(idxX < srcCopyWidth && idxY < srcCopyHeight) {
const unsigned int srcReadIdx = (blockIdx.y * blockDim.y + threadIdx.y) * srcJumpSize + blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int destWriteIdx =  (blockIdx.x * blockDim.x + threadIdx.y) * destJumpSize + blockIdx.y * blockDim.y + threadIdx.x;
__shared__ float smem[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];

smem[threadIdx.x][threadIdx.y] = srcStart[srcReadIdx];
__syncthreads();

destStart[destWriteIdx] = smem[threadIdx.y][threadIdx.x];
//    }
}