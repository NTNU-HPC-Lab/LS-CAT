#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddScalar(float* gData, float scalar, float* target, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
target[i] = scalar + gData[i];
}