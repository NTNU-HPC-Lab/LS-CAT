#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, const unsigned int width, const unsigned int height, const float scaleVec) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = mat[i] + scaleVec * vec[i / width];
}
}