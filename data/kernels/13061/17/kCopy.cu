#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCopy(float* srcStart, float* destStart, unsigned int copyWidth, unsigned int jumpWidth, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < numElements)
destStart[(idx / copyWidth) * jumpWidth + idx % copyWidth] = srcStart[(idx / copyWidth) * jumpWidth + idx % copyWidth];
}