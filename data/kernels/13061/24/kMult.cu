#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
//    const unsigned int idx = blockIdx.y * height + blockIdx.x * blockDim.x  + threadIdx.y*blockDim.x + threadIdx.x;
for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = a[i] * b[i];
}
}