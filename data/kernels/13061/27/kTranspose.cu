#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTranspose(float* a, float* dest, int width, int height) {
const int bx = blockIdx.x * blockDim.x;
const int by = blockIdx.y * blockDim.y;
const int tx = bx + threadIdx.x;
const int ty = by + threadIdx.y;
//    unsigned int idx = ty * width + tx;

__shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

if (tx < width && ty < height) {
smem[threadIdx.y][threadIdx.x] = a[ty * width + tx];
}
__syncthreads();

if (by + threadIdx.x < height && threadIdx.y + bx < width) {
//        idx = height * (blockIdx.x * blockDim.x + threadIdx.y) + blockIdx.y * blockDim.y + threadIdx.x;
dest[(bx + threadIdx.y) * height + by + threadIdx.x] = smem[threadIdx.x][threadIdx.y];
}
}