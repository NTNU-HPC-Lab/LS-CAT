#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(float* src, float* tgt, unsigned int srcWidth, unsigned int srcHeight, unsigned int tgtWidth, unsigned int tgtHeight) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;
//    const unsigned int numEls = tgtWidth * tgtHeight;
for (unsigned int i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
const unsigned int y = i / tgtWidth;
const unsigned int x = i % tgtWidth;
const unsigned int srcY = y % srcHeight;
const unsigned int srcX = x % srcWidth;
tgt[i] = src[srcY * srcWidth + srcX];
}
}