#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addTen(float* d, int count) {
int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;

int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

if(tid < count) {
d[tid] = d[tid] + 10;
}
}