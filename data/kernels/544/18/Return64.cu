#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Return64( unsigned long long *sum, unsigned long long *out, const unsigned long long *pIn )
{
out[threadIdx.x] = atomicAdd( &sum[threadIdx.x], *pIn );
}