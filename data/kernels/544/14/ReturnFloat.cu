#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReturnFloat( float *sum, float *out, const float *pIn )
{
out[threadIdx.x] = atomicAdd( &out[threadIdx.x], pIn[threadIdx.x] );
}