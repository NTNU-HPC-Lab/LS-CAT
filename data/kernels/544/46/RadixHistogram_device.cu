#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RadixHistogram_device( int *dptrHistogram, const int *in, size_t N, int shift, int mask )
{
for ( int i = blockIdx.x*blockDim.x+threadIdx.x;
i < N;
i += blockDim.x*gridDim.x ) {
int index = (in[i] & mask) >> shift;
atomicAdd( dptrHistogram+index, 1 );
}
#if 0
const int cBuckets = 1<<b;
__shared__ unsigned char sharedHistogram[NUM_THREADS][cBuckets];

for ( int i = blockIdx.x*blockDim.x+threadIdx.x;
i < N;
i += blockDim.x*gridDim.x ) {
int index = (in[i] & mask) >> shift;
if ( 0 == ++sharedHistogram[threadIdx.x][index] ) {
atomicAdd( dptrHistogram+index, 256 );
}
}
__syncthreads();
for ( int i = 0; i < cBuckets; i++ ) {
if ( sharedHistogram[threadIdx.x][i] ) {
atomicAdd( dptrHistogram+i, sharedHistogram[threadIdx.x][i] );
}
}
#endif
}