#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReturnFloat( float *sum, float *out, const float *pIn )
{
extern __shared__ float s[];
s[threadIdx.x] = pIn[threadIdx.x];
__syncthreads();
(void) atomicAdd( &s[threadIdx.x], *pIn );
__syncthreads();
out[threadIdx.x] = s[threadIdx.x];
}