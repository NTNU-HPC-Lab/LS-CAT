#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_to_first_index_h(float *X, int height, int width) {
int t = blockIdx.x * blockDim.x + threadIdx.x;
float tmp = 0;
if (t < width) {
for (int i = 0; i < height; i++) {
tmp += X[i * width + t];
}
X[t] = tmp;
}
}