#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_h(float *X, float *Y, int size_in) {
int t = blockIdx.x * blockDim.x + threadIdx.x;
if (t < size_in) {
Y[t] = 0.0;
if (X[t] >= 0)
Y[t] = X[t];
}
}