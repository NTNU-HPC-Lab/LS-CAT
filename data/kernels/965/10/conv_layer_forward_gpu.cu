#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_layer_forward_gpu(float *x, float *w, float *y, int h_in, int w_in, int w_out, int k, int m) {
int n, m_, h, w_, p, q;
n = blockIdx.x;   // Batch index
m_ = blockIdx.y;  // Channel index
h = threadIdx.y;  // Pixel (h, w_)
w_ = threadIdx.x; // Pixel (h, w_)
float ans = 0;    // Return value
int offset = n * (h_in * w_in);

// Load w into shared memory to speed up the data access
__shared__ float cached_w[CONV_KERNEL_SIZE][CONV_KERNEL_SIZE];
if (h < k && w_ < k) {
cached_w[h][w_] = w[m_ * (k * k) + h * k + w_];
}
__syncthreads();

// Loop over k by k kernel
if (h < w_out && w_ < w_out) {
for (p = 0; p < k; p++) {
for (q = 0; q < k; q++)
ans = ans + x[offset + (h + p) * w_in + (w_ + q)] * cached_w[p][q];
}
// Write out the return value
y[n * (m * w_out * w_out) + m_ * (w_out * w_out) + h * w_out + w_] = ans;
}
}