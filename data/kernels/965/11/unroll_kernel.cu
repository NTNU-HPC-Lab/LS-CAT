#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unroll_kernel(int h_in, int w_in, int k, float *x, float *x_unroll) {
int w_out_, h_out_, h_unroll, w_unroll_, p, q;
int t = blockIdx.x * 1024 + threadIdx.x; // Index of this thread
int w_out = w_in - k + 1;                // Output image size
int w_unroll = w_out * w_out;            // Unroll limit

if (t < w_unroll) {
h_out_ = t / w_out;                  // Output height
w_out_ = t % w_out;                  // Output width
w_unroll_ = h_out_ * w_out + w_out_; // The index of output pixel in image
for (p = 0; p < k; p++)
for (q = 0; q < k; q++) {
h_unroll = p * k + q;
if ((h_out_ + p) < h_in && (w_out_ + q) < w_in)
x_unroll[h_unroll * w_unroll + w_unroll_] =
x[(h_out_ + p) * w_in + w_out_ + q];
}
}
}