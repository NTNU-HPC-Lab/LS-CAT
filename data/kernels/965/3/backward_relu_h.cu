#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_relu_h(float *X, float *Y, int size_in) {
int t = blockIdx.x * blockDim.x + threadIdx.x;
if (t < size_in) {
X[t] = 0.0;
if (X[t] >= 0)
X[t] = Y[t];
}
}