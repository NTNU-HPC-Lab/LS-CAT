#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grad_descent(float *odata, const float *idata, int size) {
int t = blockIdx.x * blockDim.x + threadIdx.x;
if (t < size) {
odata[t] -= LEARNIG_RATE * idata[t];
}
}