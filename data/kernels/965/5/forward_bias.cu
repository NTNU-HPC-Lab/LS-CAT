#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_bias(float *X, float *b, int N, int ch_in, int h_in, int w_in) {
int n = blockIdx.x;
int ch = blockIdx.y;
int h = threadIdx.x;
int w = threadIdx.y;
X[n * ch_in * h_in * w_in + ch * h_in * w_in + h * w_in + w] += b[ch];
}