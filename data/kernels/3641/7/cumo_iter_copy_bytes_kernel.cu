#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cumo_iter_copy_bytes_kernel(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
char *p1_ = NULL;
char *p2_ = NULL;
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
p1_ = p1 + (idx1 ? idx1[i] : i * s1);
p2_ = p2 + (idx2 ? idx2[i] : i * s2);
memcpy(p2_, p1_, elmsz);
}
}