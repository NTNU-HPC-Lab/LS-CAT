#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cumo_na_diagonal_index_stride_kernel(size_t *idx, size_t *idx0, ssize_t s1, size_t k0, size_t k1, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = idx0[i+k0] + s1*(i+k1);
}
}