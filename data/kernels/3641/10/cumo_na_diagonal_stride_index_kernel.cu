#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cumo_na_diagonal_stride_index_kernel(size_t *idx, ssize_t s0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = s0*(i+k0) + idx1[i+k1];
}
}