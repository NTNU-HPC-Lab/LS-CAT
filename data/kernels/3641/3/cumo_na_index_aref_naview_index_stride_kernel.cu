#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = idx[i] * s1;
}
}