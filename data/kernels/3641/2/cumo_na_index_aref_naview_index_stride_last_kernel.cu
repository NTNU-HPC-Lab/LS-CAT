#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_stride_last_kernel(size_t *idx, ssize_t s1, size_t last, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = (last - idx[i]) * s1;
}
}