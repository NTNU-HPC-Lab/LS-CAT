#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_index_kernel(size_t *idx, size_t *idx1, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = idx1[idx[i]];
}
}