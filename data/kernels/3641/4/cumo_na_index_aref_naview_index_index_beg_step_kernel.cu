#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_index_beg_step_kernel(size_t *idx, size_t *idx1, size_t beg, ssize_t step, uint64_t n)
{
for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
idx[i] = idx1[beg + step * i];
}
}