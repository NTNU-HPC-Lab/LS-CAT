#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
/* Each thread gets same seed, a different sequence number,
no offset */
hiprand_init(seed, tidx, 0, &state[tidx]);
}