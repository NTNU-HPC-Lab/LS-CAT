#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] + 2*b[i];
}

}