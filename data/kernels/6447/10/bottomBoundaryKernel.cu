#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bottomBoundaryKernel(double* temperature, int block_size) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < block_size) {
temperature[(block_size + 2) * (block_size + 1) + (1 + i)] = 1.0;
}
}