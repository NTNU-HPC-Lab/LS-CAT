#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leftBoundaryKernel(double* temperature, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
temperature[(block_size + 2) * (1 + j)] = 1.0;
}
}