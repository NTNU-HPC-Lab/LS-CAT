#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initKernel(double* temperature, int block_size) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

if (i < block_size + 2 && j < block_size + 2) {
temperature[(block_size + 2) * j + i] = 0.0;
}
}