#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobiKernel(double* temperature, double* new_temperature, int block_size) {
int i = (blockDim.x * blockIdx.x + threadIdx.x) + 1;
int j = (blockDim.y * blockIdx.y + threadIdx.y) + 1;

if (i <= block_size && j <= block_size) {
new_temperature[j * (block_size + 2) + i] =
(temperature[j * (block_size + 2) + (i - 1)] +
temperature[j * (block_size + 2) + (i + 1)] +
temperature[(j - 1) * (block_size + 2) + i] +
temperature[(j + 1) * (block_size + 2) + i] +
temperature[j * (block_size + 2) + i]) *
DIVIDEBY5;
}
}