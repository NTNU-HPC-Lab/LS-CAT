#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leftPackingKernel(double* temperature, double* ghost, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
ghost[j] = temperature[(block_size + 2) * (1 + j) + 1];
}
}