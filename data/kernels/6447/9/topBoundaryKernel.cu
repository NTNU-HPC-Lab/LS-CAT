#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void topBoundaryKernel(double* temperature, int block_size) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < block_size) {
temperature[1 + i] = 1.0;
}
}