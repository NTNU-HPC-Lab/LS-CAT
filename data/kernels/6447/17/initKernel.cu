#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initKernel(double* data, int count, double val) {
int ti = blockDim.x * blockIdx.x + threadIdx.x;

if (ti < count) {
data[ti] = val;
}
}