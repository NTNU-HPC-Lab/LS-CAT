#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rightPackingKernel(double* temperature, double* ghost, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
ghost[j] = temperature[(block_size + 2) * (1 + j) + (block_size)];
}
}