#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* C, float* A, float* B, int n) {
// Get our global thread ID
int id = blockIdx.x * blockDim.x + threadIdx.x;

// Make sure we do not go out of bounds
if (id < n) {
C[id] = A[id] + B[id];
}
}