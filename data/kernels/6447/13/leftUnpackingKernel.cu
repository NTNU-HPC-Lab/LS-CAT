#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leftUnpackingKernel(double* temperature, double* ghost, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
temperature[(block_size + 2) * (1 + j) + 1] = ghost[j];
}
}