#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rightUnpackingKernel(double* temperature, double* ghost, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
temperature[(block_size + 2) * (1 + j) + (block_size)] = ghost[j];
}
}