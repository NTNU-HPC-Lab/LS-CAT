#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rightBoundaryKernel(double* temperature, int block_size) {
int j = blockDim.x * blockIdx.x + threadIdx.x;
if (j < block_size) {
temperature[(block_size + 2) * (1 + j) + (block_size + 1)] = 1.0;
}
}