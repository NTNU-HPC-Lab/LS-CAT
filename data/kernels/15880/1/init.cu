#include "hip/hip_runtime.h"
#include "includes.h"
///
/// Copyright (c) 2018, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    dgemm
///
/// PURPOSE: This program tests the efficiency with which a dense matrix
///          dense multiplication is carried out
///
/// USAGE:   The program takes as input the matrix order,
///          the number of times the matrix-matrix multiplication
///          is carried out, and, optionally, a tile size for matrix
///          blocking
///
///          <progname> <# iterations> <matrix order>
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// FUNCTIONS CALLED:
///
///          Other than OpenMP or standard C functions, the following
///          functions are used in this program:
///
///          cblasDgemm()
///
/// HISTORY: Written by Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, December, 2017.
///
//////////////////////////////////////////////////////////////////////




__global__ void init(int order, double * C)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

if ((i<order) && (j<order)) {
C[i*order+j] = 0;
}
}