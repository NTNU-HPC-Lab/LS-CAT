#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
__shared__ float minvals[128];
__shared__ float maxvals[128];
const int tx = threadIdx.x;
const int x = __mul24(blockIdx.x, 128) + tx;
const int y = __mul24(blockIdx.y, 16);
const int b = blockDim.x;
int p = __mul24(y, width) + x;
if (x<width) {
float val = d_Data[p];
minvals[tx] = val;
maxvals[tx] = val;
} else {
float val = d_Data[p-x];
minvals[tx] = val;
maxvals[tx] = val;
}
for (int ty=1;ty<16;ty++) {
p += width;
if (tx<width) {
float val = d_Data[p];
if (val<minvals[tx])
minvals[tx] = val;
if (val>maxvals[tx])
maxvals[tx] = val;
}
}
__syncthreads();
int mod = 1;
for (int d=1;d<b;d<<=1) {
if ((tx&mod)==0) {
if (minvals[tx+d]<minvals[tx+0])
minvals[tx+0] = minvals[tx+d];
if (maxvals[tx+d]>maxvals[tx+0])
maxvals[tx+0] = maxvals[tx+d];
}
mod = 2*mod + 1;
__syncthreads();
}
if (tx==0) {
int ptr = 2*(__mul24(gridDim.x,blockIdx.y) + blockIdx.x);
d_MinMax[ptr+0] = minvals[0];
d_MinMax[ptr+1] = maxvals[0];
}
__syncthreads();
}