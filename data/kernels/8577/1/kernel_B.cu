#include "hip/hip_runtime.h"
#include "includes.h"





__global__ void kernel_B( float *g_data, int dimx, int dimy )
{
int id  = blockIdx.x*blockDim.x + threadIdx.x;

float value = g_data[id];

if( id % 2 )
{
value += sqrtf( logf(value) + 1.f );
}
else
{
value += sqrtf( cosf(value) + 1.f );
}

g_data[id] = value;
}