#include "hip/hip_runtime.h"
#include "includes.h"





__global__ void kernel_D( float * _g_data, int dimx, int dimy )
{
float4* g_data = reinterpret_cast<float4 *>(_g_data);

int id  = blockIdx.x*blockDim.x + threadIdx.x;

float4 value = g_data[id];

value.x += sqrtf( cosf(value.x) + 1.f );
value.y += sqrtf( logf(value.y) + 1.f );
value.z += sqrtf( cosf(value.z) + 1.f );
value.w += sqrtf( logf(value.w) + 1.f );

g_data[id] = value;
}