#include "hip/hip_runtime.h"
#include "includes.h"





__global__ void kernel_A( float *g_data, int dimx, int dimy )
{
int ix  = blockIdx.x;
int iy  = blockIdx.y*blockDim.y + threadIdx.y;
int idx = iy*dimx + ix;

float value = g_data[idx];

if( ix % 2 )
{
value += sqrtf( logf(value) + 1.f );
}
else
{
value += sqrtf( cosf(value) + 1.f );
}

g_data[idx] = value;
}