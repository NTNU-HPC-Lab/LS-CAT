#include "hip/hip_runtime.h"
#include "includes.h"





__global__ void kernel_C( float * _g_data, int dimx, int dimy )
{
float2* g_data = reinterpret_cast<float2 *>(_g_data);

int id  = blockIdx.x*blockDim.x + threadIdx.x;

float2 value = g_data[id];

value.x += sqrtf( cosf(value.x) + 1.f );
value.y += sqrtf( logf(value.y) + 1.f );

g_data[id] = value;
}