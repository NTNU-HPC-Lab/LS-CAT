#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
float val = 1.0f;
float cur_weight = *cur_tot_weight;

if (max_count > 0.0f) {
float new_scale = max((float)cur_weight, (float)max_count) / max_count;

float prior_scale_change = new_scale - 1.0f;
val += prior_scale_change;
}

for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
i += blockDim.x * gridDim.x) {
int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
quadratic[diag_idx] += val;
}

if (threadIdx.x == 0) {
linear[0] += val * prior_offset;
}
}