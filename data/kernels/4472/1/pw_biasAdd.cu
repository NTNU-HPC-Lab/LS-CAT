#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) y[i] += bias[i % nBias];
}