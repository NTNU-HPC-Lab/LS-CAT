#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) y[i] = a[i] + b[i];
}