#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_tanh(float *y, float *a, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) y[i] = tanh(a[i]);
}