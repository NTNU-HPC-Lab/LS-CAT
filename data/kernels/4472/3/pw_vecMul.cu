#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) y[i] = a[i] * b[i];
}