#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaive(float *odata, const float *idata)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
odata[x*width + (y+j)] = idata[(y+j)*width + x];
}