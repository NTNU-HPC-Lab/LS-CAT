#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void spinKernel(unsigned long long timeout_clocks = 100000ULL)
{
register unsigned long long start_time, sample_time;
start_time = clock64();
while(1) {
sample_time = clock64();
if (timeout_clocks != ~0ULL && (sample_time - start_time) > timeout_clocks) {
break;
}
}
}