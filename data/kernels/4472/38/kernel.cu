#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *a, int offset)
{
int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
float x = (float)i;
float s = sinf(x);
float c = cosf(x);
a[i] = a[i] + sqrtf(s*s+c*c);
}