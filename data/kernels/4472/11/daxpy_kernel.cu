#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void daxpy_kernel(int n, double a, double * x, double * y)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n)
{
y[i] = a*x[i] + y[i];
}
}