#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_data_kernel( int n, double* x)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if ( i < n )
{
x[i] = n - i;
}
}