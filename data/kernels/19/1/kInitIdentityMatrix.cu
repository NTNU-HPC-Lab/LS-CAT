#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kInitIdentityMatrix(float* a, int size, int num_elements) {
const int idxX = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
for (int x = idxX; x < num_elements; x += gridDim.x * THREADS_PER_BLOCK) {
if (x % size == x / size) {
a[x] = 1;
} else {
a[x] = 0;
}
}
}