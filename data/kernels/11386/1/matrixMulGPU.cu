#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulGPU( int * a, int * b, int * c )
{
int val = 0;

int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if (row < N && col < N)
{
for ( int k = 0; k < N; ++k )
val += a[row * N + k] * b[k * N + col];
c[row * N + col] = val;
}
}