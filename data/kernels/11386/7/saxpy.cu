#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(float scalar, float * x, float * y)
{
// Determine our unique global thread ID, so we know which element to process
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if ( tid < N ) // Make sure we don't do more work than we have data!
y[tid] = scalar * x[tid] + y[tid];
}