#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deviceAddVector(int *d_a, int *d_b, int *d_c, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) {
d_c[i] = d_a[i] + d_b[i];
//  printf("Tread %d make sum %d + %d = %d", i, d_a[i], d_b[i], d_c[i]);
}
}