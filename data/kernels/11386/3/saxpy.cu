#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
// Determine our unique global thread ID, so we know which element to process
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if ( tid < N ) // Make sure we don't do more work than we have data!
c[tid] = 2 * a[tid] + b[tid];
}