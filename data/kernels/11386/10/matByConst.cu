#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matByConst(unsigned char *img, unsigned char *result, int alpha, int cols, int rows) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if (row < rows && col < cols) {
int idx = row * cols + col;
result[idx] = img[idx] * alpha;
}
}