#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy2D(float scalar, float * x, float * y)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if ( row < NX && col < NY ) // Make sure we don't do more work than we have data!
y[row*NY+col] = scalar * x[row*NY+col] + y[row*NY+col];
}