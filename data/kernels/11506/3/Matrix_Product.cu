#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_Product (double *A, double *g, double *C)
// Each thread computes one element of C
// by accumulating results into Cvalue
{               double Cvalue = 0.00;
int row = blockIdx.y*blockDim.y+threadIdx.y;
// int col = blockIdx.x * blockDim.x + threadIdx.x;
//size of matrix A//
int N=1000;
if(row> N ) return;
for (int e = 0; e < N; e++)
{
Cvalue += A[N*row+e]*g[e];
}
C[row]+= Cvalue;
}