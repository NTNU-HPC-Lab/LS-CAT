#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Product (float *a, float *b, float *c)
{
// Out of all the threads created each one computes 1 value of C and stores into cval

float cval = 0.00;
int R = blockIdx.y * blockDim.y + threadIdx.y; //Row of the matrix
int C = blockIdx.x * blockDim.x + threadIdx.x; //Column of the matrix
//Defining the size of the matrix//
int N=1000;
if(R> N || C > N ){
return;
}
for (int j = 0; j < N; j++)
{
cval += a[R * N+ j] *b[j * N + C];

}
c[R * N + C]+= cval;
}