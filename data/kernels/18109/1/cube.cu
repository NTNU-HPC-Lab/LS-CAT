#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube(double* d_out, double* d_in)
{
int idx = threadIdx.x;
double f = d_in[idx];
d_out[idx] = f*f*f;

}