#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void calc2points(float* point_coordinate_1, float* point_coordinate_2 , float* coordinates_arr)
{
int tid = threadIdx.x; // 52

coordinates_arr[tid] = pow(point_coordinate_1[tid] - point_coordinate_2[tid],2);
}