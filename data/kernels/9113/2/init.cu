#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int *arr, int sqroot, int limit) {
int c;
for(c = 2; c <= sqroot; c++) {
if(arr[c] == 0) {
/*
#pragma omp parallel for shared(arr, limit, c) private(m)
for(m = c+1; m < limit; m++) {
if(m%c == 0) {
arr[m] = 1;
}
}
*/
int tid = c+1+ threadIdx.x + (blockIdx.x * blockDim.x);
if (tid<limit){
if (tid % c ==0) {
arr[tid] = 1;
}
}


}
}
}