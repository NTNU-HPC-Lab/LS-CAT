#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul_tileKernel(float* Md, float* Nd, float* Pd, int Width){

int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;
int tx = threadIdx.x, ty = threadIdx.y;
__shared__ float a[TILE_WIDTH][TILE_WIDTH], b[TILE_WIDTH][TILE_WIDTH];
float Pvalue = 0;
//Each thread computes one element of the block sub-matrix
for(int k=0; k< Width/TILE_WIDTH; k++){
a[ty][tx] = Md[Row*Width+k*TILE_WIDTH+tx];
b[ty][tx] = Nd[Col+Width*(k*TILE_WIDTH + ty)];
__syncthreads(); //sync all threads in a block;
for(int kk=0; kk<TILE_WIDTH; kk++)
Pvalue += a[ty][kk]*b[kk][tx];
__syncthreads(); //avoid memory hazards;
}
Pd[Row*Width+Col] = Pvalue;
}