#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult(float *A, float *B, float *C,  int n)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if (row < n && col < n) {
for (int i = 0; i < n; ++i) {
C[row * n + col] += A[row * n + i] * B[i * n + col];
}
}
}