#include "hip/hip_runtime.h"
#include "includes.h"
/* Matrix normalization.
* Compile with "nvcc matrixNormCuda.c -lm"
*/


/* Program Parameters */
#define N 8000  /* Matrix size */
int blocks_per_grid = 32;
int threads_per_block = 256;

/* Matrices */
float A[N*N], B[N*N];

/* CUDA arrays */
float *A_d, *B_d;


/* Initialize A and B*/
__global__ void matrixNorm(float* A_dd, float* B_dd, int N_d) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// __shared__ float mu, sigma;
float mu, sigma;
int row;

if (idx < N_d) {
mu = 0.0;
for (row=0; row < N_d; row++){
mu += A_dd[row*N_d + idx];
}
mu /= N_d;

sigma = 0.0;
for (row=0; row < N_d; row++){
sigma += powf(A_dd[row*N_d + idx] - mu, 2.0);
}
sigma /= N_d;
sigma = sqrt(sigma);

for (row=0; row < N_d; row++) {
if (sigma == 0.0){
B_dd[row*N_d + idx] = 0.0;
}
else{
B_dd[row*N_d + idx] = (A_dd[row*N_d + idx] - mu) / sigma;
}
}
}
}