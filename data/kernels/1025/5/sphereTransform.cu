#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sphereTransform(float *data, const unsigned int N)
{
unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
if (idx < N)
{
data[idx] = data[idx] * 360.0f - 180.0f;
data[idx + N] = acosf(2.0f * data[idx + N] - 1.0f);
}
}