#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply( float *A2, float *B2, float *C, int N, int threads_num ){
__shared__ float *A;
__shared__ float *B;
A = A2;	B = B2;

float tmp;
int k, pos;

int a = N * N * (blockDim.x * blockIdx.x + threadIdx.x) / threads_num, b;

if ( blockDim.x * blockIdx.x + threadIdx.x == threads_num - 1)
b = N * N;
else
b = N * N * ( blockDim.x * blockIdx.x + threadIdx.x + 1) / threads_num;

for( pos = a; pos < b; pos++ ){
tmp = 0;
for( k = 0; k < N; k++ )
tmp += A[ N * (pos / N ) + k ] * B[ k * N + pos - ( pos / N) * N];
C[ pos ] = tmp;
}
}