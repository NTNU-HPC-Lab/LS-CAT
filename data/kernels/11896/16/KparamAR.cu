#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void  KparamAR(double* a, double* b, double* c, double ss, double dtAR, int width, int height)
{
#define eps 1E-12;

int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if (i >= width || j >= height) return;

int  x= i;
int  y= j;

if (i> width/2)  x = width-i;
if (j> height/2) y = height-j;

double r = sqrt( (double)x*x + (double)y*y )+Eps;

a[i+j*width] =  2-dtAR*2*ss*r- pow(dtAR*ss*r,2);
b[i+j*width] = -1+dtAR*2*ss*r;
// c[i+j*width] =  50* pow(dtAR,2);
// Correction Jonathan 7-12-16
c[i+j*width] =  1;

}