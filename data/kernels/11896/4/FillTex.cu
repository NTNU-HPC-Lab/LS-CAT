#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void FillTex(void *surface, int width, int height, size_t pitch, double* src, int Mask)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

unsigned char *pixel1;

if (x >= width || y >= height) return;

double w = src[x + width*y];

if (w<0) {w=0;}
if (w>253) {w=253;}


pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;

//pixel1[3] = 255;                     // alpha = 255 sauf s'il fait partie du masque
for (int i=0;i<4;i++)
{  if (Mask & (1<<i))  pixel1[i] = w;  }
}