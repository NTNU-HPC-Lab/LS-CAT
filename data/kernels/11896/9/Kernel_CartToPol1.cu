#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void Kernel_CartToPol1(double *tb1, double *tb2, int width, int height )
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x >= width || y >= height) return;

double R = 2* sqrtf( powf(x-width/2,2) + powf(y-height/2,2) );
double theta = (atan2f( y-height/2 ,  x-width/2) +PI)*height/(2*PI);

int x1 = ((int) R) % width ;
int y1 = ((int) theta) % height;

int xp1 = (x1+1) % width;
int yp1 = (y1+1) % height;

double z1 =   tb1[width*y1+x1];
double z2 =   tb1[width*yp1+x1];
double z3 =   tb1[width*yp1+xp1];
double z4 =   tb1[width*y1+xp1];

double dx =  theta-floorf(theta);
double dy =  R-floorf(R);

double zp = 1.0*z1+ dy*(1.0*z2-z1);
double zq = 1.0*z4+ dy*(1.0*z3-z4);
double ZR = zp+ dx*(zq-zp);

tb2[width*y+x] = ZR;

}