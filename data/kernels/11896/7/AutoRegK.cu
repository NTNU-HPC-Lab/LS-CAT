#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void AutoRegK(double* X0, double* X1, double* X2, double* N0, int numElements, double a, double b)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
X2[i] = a*X1[i] + b*X0[i] + N0[i];
}
}