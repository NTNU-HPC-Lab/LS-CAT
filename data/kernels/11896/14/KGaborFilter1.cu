#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void KGaborFilter1(double* filter, double* Vr, int width, int height, double ss , double r0, double sr0, double stheta0 )
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if (i >= width || j >= height) return;

double x = i;
double y = j;

if (i> width/2)  x = width-i;
if (j> height/2) y = height-j;

#define Eps 1E-6;
double r = sqrt(x*x+ y*y)+Eps;

double theta;
if (x>0) theta= atan2( y, x); else theta = PI/2;

//double ff =  exp( cos(2*theta)/stheta0 )
//             *
//             exp(-0.5*pow(log(r/r0),2)/log(1+pow(sr0,2))) * pow(r0/r,3)*ss*r;

// Correction Jonathan 7-12-16
double ff =  exp( cos(2*theta)/(4*pow(stheta0,2) ) )
*
exp(-0.5*pow(log(r/r0),2)/log(1+pow(sr0,2))) * pow(r0/r,3)*4*pow(ss*r,3);


filter[i+j*width] = ff;
if (i>0 || j>0) Vr[i+j*width] =  ff/(4*pow(ss*r,3)); else Vr[i+j*width] = 0;


}