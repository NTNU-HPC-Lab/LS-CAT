#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void AutoRegPinkK(double* X0, double* X1, double* Y, double* C0, double* D0, double* LastF, int numElts, int Nc)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElts)
{
for (int j=0; j<Nc; j++)
LastF[j*numElts+i] = (X0[i]+X1[i])*D0[j] - C0[j]*LastF[j*numElts+i];

double w = 0;
for (int j=0; j<Nc; j++) w = w+ LastF[j*numElts+i];
Y[i] = w;
}
}