#include "hip/hip_runtime.h"
#include "includes.h"


// includes, project

#define PI 3.1415926536f


int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;


// Conversion d'un vecteur réel en vecteur complexe

// Conversion d'un vecteur complexe en vecteur réel


// Multiplie point par point un vecteur complex par un vecteur réel

// Applique y = at*x +bt à chaque point d'un vecteur réel




// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha n'est pas modifié

// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de bytes
// Alpha n'est pas modifié


// Remplissage de la linearmem (tableau de pixels) associée à la texture avec le tableau de réel
// Alpha autorise l'affichage au dessus d'un certain seuil


// Processus auto-régressif X2 = a*X1 + b*X0 + N0;



// Expansion
// On applique une interpolation bi-linéaire à la source

// Transformation Cartesian To Polar
// On applique une interpolation bi-linéaire à la source



__global__ void KtexFillRect(void* surface, double* tb, int width, int height, size_t pitch, float2* Pts, int k, float th)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

unsigned char *pixel1;

if (x >= width || y >= height) return;

pixel1 = (unsigned char *)( (char*)surface + y*pitch) + 4*x;

if (
((Pts[1].y-Pts[0].y)*(x-Pts[0].x)-( y-Pts[0].y)*(Pts[1].x-Pts[0].x)>=0)
&&
((Pts[2].y-Pts[1].y)*(x-Pts[1].x)-( y-Pts[1].y)*(Pts[2].x-Pts[1].x)>=0)
&&
((Pts[3].y-Pts[2].y)*(x-Pts[2].x)-( y-Pts[2].y)*(Pts[3].x-Pts[2].x)>=0)
&&
((Pts[0].y-Pts[3].y)*(x-Pts[3].x)-( y-Pts[3].y)*(Pts[0].x-Pts[3].x)>=0)
&&
(pixel1[k]>=th)
)
tb[x + width*y] = 1;



}