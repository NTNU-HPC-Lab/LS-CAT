#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printMat( const double *A, int size )
{
if( threadIdx.x == 0 && blockIdx.x == 0 )
for( int i = 0; i < size; i++ )
printf("A[%d] = %f\n",i,A[i]);
return;
} /* end printMat */