#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce( float *a, int size, int c) {
int tid = blockIdx.x;	//Handle the data at the index
int index=c,j=0;//size=b

for(j=index+1;j<size;j++) {
a[((tid+index+1)*size + j)] = (float)(a[((tid+index+1)*size + j)] - (float)a[((tid+index+1)*size+index)] * a[((index*size) + j)]);
}

}