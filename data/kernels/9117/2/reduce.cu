#include "hip/hip_runtime.h"
#include "includes.h"

float *A,*L,*U,*input;
void arrayInit(int n);
void verifyLU(int n);
void updateLU(int n);
void freemem(int n);

/*
*/


__global__ void reduce( float *a, int size, int c) {
int tid = blockIdx.x;	//Handle the data at the index
int thid = threadIdx.x;
int index=c,j=0;//size=b
int numthreads = blockDim.x;
for(j=index+1;j<size;j+=numthreads) {
a[((tid+index+1)*size + j+thid)] = (float)(a[((tid+index+1)*size + j+thid)] - (float)a[((tid+index+1)*size+index)] * a[((index*size) + j+thid)]);
}

}