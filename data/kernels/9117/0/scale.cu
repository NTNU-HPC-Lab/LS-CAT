#include "hip/hip_runtime.h"
#include "includes.h"

float *A,*L,*U,*input;
void arrayInit(int n);
void verifyLU(int n);
void updateLU(int n);
void freemem(int n);

/*
*/


__global__ void scale( float *a, int b, int c) {
int index=c,size=b,k=0;

for(k=index+1;k<size;k++) {
a[size*index + k] = a[size*index + k] / a[size*index + index];
}

}