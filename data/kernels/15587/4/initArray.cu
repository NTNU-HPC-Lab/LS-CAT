#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initArray(uint32_t * path, double *approx, uint32_t *top_k, int n){
int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < n){
for(int i = 0; i < sizeof(path); i++){
approx[i]++;
top_k[i] = path[i]++;
}
}
}