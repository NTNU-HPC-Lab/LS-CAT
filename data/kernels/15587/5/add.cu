#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int n){
int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < n)
c[index] = a[index] + b[index];
}