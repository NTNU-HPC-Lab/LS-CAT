#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void do_scale(uint8_t * inBuffer, uint8_t * outBuffer, uint32_t inWidth, uint32_t inHeight, uint32_t outWidth, uint32_t outHeight, double ratioHeight, double ratioWidth, double xRatio, double yRatio, uint32_t outHeightOffset, uint32_t outWidthOffset, uint32_t inStep, uint32_t outStep)
{
uint32_t outRowIndex=blockIdx.x+outHeightOffset;
uint32_t outColIndex=threadIdx.x+outWidthOffset;
uint32_t outIndex=(outRowIndex)*outWidth+outColIndex;

uint32_t inX00row=outRowIndex/ratioHeight;
uint32_t inX00col=outColIndex/ratioWidth;
//if (inX00row >= inHeight-2){
//printf("==edge inX00row[%u]inHeight[%u]==\n", inX00row, inHeight);
//inX00row=inHeight-2;
//}
//if (inX00col >= inWidth-2){
//printf("==edge inX00col[%u]inWidth[%u]==\n", inX00col, inWidth);
//inX00col=inWidth-2;
//}
uint8_t inX00=inBuffer[(inX00row*inWidth+inX00col)*inStep];
uint8_t inX01=inBuffer[(inX00row*inWidth+inX00col+1)*inStep];
uint8_t inX10=inBuffer[((inX00row+1)*inWidth+inX00col)*inStep];
uint8_t inX11=inBuffer[((inX00row+1)*inWidth+inX00col+1)*inStep];
outBuffer[outIndex*outStep]=yRatio*(xRatio*(inX00)+(1-xRatio)*(inX01))+(1-yRatio)*(xRatio*(inX10)+(1-xRatio)*(inX11));
}