#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
int i=blockIdx.x*COL+threadIdx.x;
//printf("[gpu]:gridDim.x=%u, gridDim.y=%u, gridDim.z=%u, blockDim.x=%u, blockDim.y=%u, blockDim.z=%u, blockIdx.x=%u, blockIdx.y=%u, blockIdx.z=%u,threadIdx.x=%u, threadIdx.y=%u, threadIdx.z=%u\n",
//gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z, blockIdx.x, blockIdx.y, blockIdx.z,threadIdx.x, threadIdx.y, threadIdx.z);
C[i]=A[i]+B[i];
//printf("sum[%u][%u]: A[%5.5f]+B[%5.5f]=C[%5.5f]\n",blockIdx.x, threadIdx.x, A[i], B[i], C[i]);
}