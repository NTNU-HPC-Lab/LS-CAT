#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
if(index < length) {
out[indices[index]] = values[index];
}
}
}