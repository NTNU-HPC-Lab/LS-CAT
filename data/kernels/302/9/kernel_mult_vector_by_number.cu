#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_mult_vector_by_number(double      *vec, double       alpha, int numElements)
{
int iam = threadIdx.x;
int bid = blockIdx.x;
int threads_in_block = blockDim.x;
int gid = bid*threads_in_block + iam;

if (gid < numElements){
vec[gid] *= alpha;
}

}