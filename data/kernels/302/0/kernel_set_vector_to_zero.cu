#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_set_vector_to_zero(double *d_vec, int dimension)
{

int iam = threadIdx.x;
int bid = blockIdx.x;
int threads_in_block = blockDim.x;
int gid = bid*threads_in_block + iam;

if (gid < dimension){
d_vec[gid] = 0;
}
}