#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_vec_equals_vec1_plus_alpha_times_vec2(double      *vec, double      *vec1, double       alpha, double      *d_a1, double      *vec2, int numElements)
{
int iam = threadIdx.x;
int bid = blockIdx.x;
int threads_in_block = blockDim.x;
int gid = bid*threads_in_block + iam;

if (gid < numElements){
double a = alpha;
if (d_a1) a *= *d_a1;

vec[gid] = vec1[gid] + a * vec2[gid];
}
}