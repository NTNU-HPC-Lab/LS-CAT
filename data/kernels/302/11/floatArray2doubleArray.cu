#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floatArray2doubleArray(const float * floatArray, double* doubleArray, const int size) {
int i = blockDim.x*blockIdx.x + threadIdx.x;
if (i < size) {
doubleArray[i] = (double) floatArray[i];
}
}