#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_add_regularization_term(double     * d_input_vector, int           dimension, double       regularization_parameter, double     * d_rv)
{
if (threadIdx.x == 0) {
double sum = 0;
for (int i = 1; i < dimension; ++i) {
sum += 0.5 * d_input_vector[i] * d_input_vector[i] * regularization_parameter;
}
*d_rv += sum;
}
}