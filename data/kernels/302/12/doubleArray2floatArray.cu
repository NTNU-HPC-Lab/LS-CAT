#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void doubleArray2floatArray(const double * doubleArray, float* floatArray, const int size) {
int i = blockDim.x*blockIdx.x + threadIdx.x;
if (i < size) {
floatArray[i] = (float) doubleArray[i];
}
}