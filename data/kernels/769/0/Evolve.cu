#include "hip/hip_runtime.h"
#include "includes.h"



#define BLOCK_SIZE 16




__global__ void Evolve(bool* field, float* scores, double b, int size, bool* next_field)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int memberIndex;

// Score
if (col >= size || row >= size)
return;

//printf("(%i, %i)\n", col, row);

float score = 0;

for (int i = -1; i <= 1; i++) //Row
{
for (int j = -1; j <= 1; j++) //Col
{
memberIndex = (col + i + size) % size + size * ((row + j + size) % size);

if (field[memberIndex] == true)
score++;
}
}

if (!field[row*size + col])
scores[row*size + col] = score * b;
else
scores[row*size + col] = score;


__syncthreads();


// Strategy
int bestStrategyIndex = row*size + col;

for (int i = -1; i <= 1; i++) //Row
{
for (int j = -1; j <= 1; j++) //Col
{
memberIndex = (col + i + size) % size + size * ((row + j + size) % size);

if (scores[bestStrategyIndex] < scores[memberIndex])
{
bestStrategyIndex = memberIndex;
}
}
}

next_field[row*size + col] = field[bestStrategyIndex];

__syncthreads();
}