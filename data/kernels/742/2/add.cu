#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{

int i = (blockIdx.x*blockDim.x) + threadIdx.x;

gc[5120 * 6 + i * 6    ] = b[i] * 0.00390625;
//gc[5120 * 6 + i * 6    ] = float(b[i]) / 256;
gc[5120 * 6 + i * 6 + 1] = g[i] * 0.00390625;
//gc[5120 * 6 + i * 6 + 1] = float(g[i]) / 256;
gc[5120 * 6 + i * 6 + 2] = r[i] * 0.00390625;
//gc[5120 * 6 + i * 6 + 2] = float(r[i]) / 256;

//	gc[5120 * 6 + i * 6 + 3] = float(i - ((i>>9)<<9) );  // i%512
//gc[5120 * 6 + i * 6 + 3] = float(i % 512);
//	gc[5120 * 6 + i * 6 + 4] = float( i >> 9);
//gc[5120 * 6 + i * 6 + 4] = float((i - (i % 512)) / 512);
//	gc[5120 * 6 + i * 6 + 5] = float(a[i]);
}