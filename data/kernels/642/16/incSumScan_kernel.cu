#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void incSumScan_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals)
{
unsigned int tIdx = threadIdx.x;
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
extern __shared__  unsigned int s_incScan[];
if (gIdx >= numVals) return;

s_incScan[tIdx] = d_inVals[tIdx];
__syncthreads();

for (int offset = 1; offset <= numVals; offset = offset * 2)
{
unsigned int temp = s_incScan[tIdx];
unsigned int neighbor = 0;
if (tIdx >= offset ) {
neighbor = s_incScan[tIdx - offset];
__syncthreads();
s_incScan[tIdx] = temp + neighbor;
}
__syncthreads();
}
d_outVals[tIdx] = s_incScan[tIdx];
}