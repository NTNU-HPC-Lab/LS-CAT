#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getPredicate_kernel(unsigned int * d_inVal, unsigned int * d_predVal, unsigned int numElems, unsigned int bitMask)
{

unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;

if (gIdx < numElems)
{
// if bitmask matches inputvale then assign 1 to the position otherwise set to 0
// we'll need to run an inclusive scan later to get the position
d_predVal[gIdx] = ((d_inVal[gIdx] & bitMask) == bitMask) ? 1 : 0;
//d_npredVal[gIdx] = ((d_inVal[gIdx] & bitMask) == bitMask) ? 0 : 1;
}
}