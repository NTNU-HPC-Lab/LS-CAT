#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverseSort_kernel(unsigned int * d_newArray, unsigned int * d_oldArray, unsigned int numElems)
{
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
if (gIdx < numElems)
{
d_newArray[gIdx] = d_oldArray[(numElems - 1)- gIdx];
}
}