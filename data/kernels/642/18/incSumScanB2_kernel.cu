#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void incSumScanB2_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals, unsigned int* d_blockOffset)
{
//	unsigned int tIdx = threadIdx.x;
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
extern __shared__  unsigned int s_incScan[];
if (gIdx >= numVals) return;

d_outVals[gIdx] = ( blockIdx.x > 0) ? d_inVals[gIdx] + d_blockOffset[blockIdx.x]: d_inVals[gIdx];

}