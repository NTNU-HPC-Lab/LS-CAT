#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swapVals_kernel(unsigned int * d_newArray, unsigned int * d_oldArray, unsigned int numElems)
{
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
if (gIdx < numElems)
{
d_newArray[gIdx] = d_oldArray[gIdx];
}
}