#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void incSumScanB1_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals, unsigned int* d_blockOffset, unsigned int valOffset)
{
unsigned int tIdx = threadIdx.x;
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
extern __shared__  unsigned int s_incScan[];
if (gIdx >= numVals) return;

//if it is the first element of a block then we need to add the offset to it.
s_incScan[tIdx] = (tIdx == 0)? d_inVals[gIdx] + valOffset: d_inVals[gIdx];

//	if (tIdx == 0) printf("gIdx =  %d,  d_inVals[ %d ] = %d , s_incScan[ %d ] = %d ,  valOffset = %d .\n", gIdx, gIdx, d_inVals[gIdx], tIdx, s_incScan[tIdx], valOffset);
__syncthreads();

//for (int offset = 1; offset <= numVals; offset = offset * 2)
for (int offset = 1; offset <= blockDim.x; offset = offset * 2)
{
unsigned int temp = s_incScan[tIdx];
unsigned int neighbor = 0;
if (tIdx >= offset) {
neighbor = s_incScan[tIdx - offset];
__syncthreads();
s_incScan[tIdx] = temp + neighbor;
}
__syncthreads();
}
d_outVals[gIdx] = s_incScan[tIdx];

//now set the cumulative sum for this block in the the blockoffsetarray
if ((tIdx + 1) == blockDim.x)
{
if ((blockIdx.x + 1) < gridDim.x)
{
d_blockOffset[blockIdx.x + 1] = s_incScan[tIdx]; //this will still need to be summed with other blocks
}
}
//	if (gIdx < 10 || gIdx > (numVals - 10)) printf("gIdx =  %d,  d_inVals[ %d ] = %d, d_outvals[ %d ] = %d , s_incScan[ %d ] = %d ,  valOffset = %d .\n",
//		 gIdx, gIdx, d_inVals[gIdx], gIdx, d_outVals[gIdx], tIdx, s_incScan[tIdx], valOffset);

}