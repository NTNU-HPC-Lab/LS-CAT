#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arraySet_kernel(unsigned int* d_vals, unsigned int value, size_t num_vals)
{
//	tIdx = threadIdx.x;
unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
if (gIdx < num_vals) d_vals[gIdx] = value;
}