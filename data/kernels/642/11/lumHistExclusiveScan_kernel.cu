#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lumHistExclusiveScan_kernel(unsigned int *d_out, unsigned int *d_in, int numItems)
{
extern __shared__ unsigned int s_exScan[];
int tid = threadIdx.x;

s_exScan[tid] = (tid > 0) ? d_in[tid - 1] : 0;
__syncthreads();

for (int offset = 1; offset <= numItems; offset = offset * 2){
unsigned int temp = s_exScan[tid];
unsigned int neighbor = 0;
if ((tid - offset) >= 0) {
neighbor = s_exScan[tid - offset];
__syncthreads();
s_exScan[tid] = temp + neighbor;
}
__syncthreads();
}
d_out[tid] = s_exScan[tid];
}