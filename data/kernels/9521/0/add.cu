#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Vector Addition - Simple addition using Cuda.
* Author - Malhar Bhatt
* Subject - High Performance Computing
*/


/** Function Add -
* Usage - Add 2 values
* Returns - Void
*/
__global__ void add( int num1, int num2, int *ans )
{
*ans = num1 + num2;
}