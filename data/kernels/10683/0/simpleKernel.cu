#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simpleKernel(float *dst, float *src1, float *src2)
{
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
//float temp = src[idx];
dst[idx] = src1[idx] + src2[idx];
}