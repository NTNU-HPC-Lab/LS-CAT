#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simpleKernel(float *dst, float *src)
{
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
float temp = src[idx];
dst[idx] = temp * temp;
}