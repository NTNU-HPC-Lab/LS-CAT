#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GRID_SIZE 32
#define SHARED_MEM 16384


__global__ void findY(float *x, float *y, int n, float h, float z, int zLoc, float *returnVal) {
// int col = blockIdx.x * blockDim.x + threadIdx.x;
// int row = blockIdx.y * blockDim.y + threadIdx.y;

__shared__ float sum;
sum = 0;
// float absVal = 0;
int count = 0;
for(int i = 0; i < n; i++) {
// absVal = abs(x[i] - z);
if(abs(x[i] - z) < h) {
//sum = atomicAdd(&sum, y[zLoc]);
sum += y[i];
// cuPrintf("sum = %d\n", sum);
count++;
}
}
*returnVal = sum / count;
// sum = 0;
// count = 0;
}