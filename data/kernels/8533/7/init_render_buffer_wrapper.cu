#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
const uint32_t step = blockDim.x * gridDim.x;
for (uint32_t i = start; i < qw_count; i += step) {
render_buffer[i] = EMPTY_KEY_64;
}
}
__global__ void init_render_buffer_wrapper(int64_t* render_buffer, const uint32_t qw_count) {
init_render_buffer(render_buffer, qw_count);
}