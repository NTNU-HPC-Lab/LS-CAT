#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
int i = blockDim.x * blockIdx.x +  threadIdx.x;
if (i < size)
{
c[i] = a[i] + b[i];
}
}