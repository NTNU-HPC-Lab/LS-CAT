#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, int threshold)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size)
{
if (image[i] > threshold)
{
moddedimage[i] = 255;
}
else
{
moddedimage[i] = 0;
}
}
}