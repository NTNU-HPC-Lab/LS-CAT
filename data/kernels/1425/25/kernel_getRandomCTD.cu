#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_getRandomCTD(double* dev_nt, double* dev_tran, double* dev_nr, double* dev_ramR, unsigned int out, int rSize, int tSize )
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
float myrand;

hiprandState s;
hiprand_init(out, tid, 0, &s);

//myrand = hiprand_uniform(&s);
//myrand *= (0 - nC);
//myrand += (nC - 0);
//dev_ramC[tid] = (int)truncf(myrand);

myrand = hiprand_uniform(&s);
myrand *= (0 - tSize);
myrand += (tSize - 0);
int t = ((int)truncf(myrand) + blockIdx.x * tSize) * 2;
//int t = (blockIdx.x * tSize) * 2;
for (int n = 0; n < 2; n++)
{
dev_tran[tid * 2 + n] = dev_nt[t + n];
}

myrand = hiprand_uniform(&s);
myrand *= (0 - rSize);
myrand += (rSize - 0);
int r = ((int)truncf(myrand) + blockIdx.x * rSize) * 4;
//int r = (blockIdx.x + blockIdx.x * rSize) * 4;
for (int n = 0; n < 4; n++)
{
dev_ramR[tid * 4 + n] = dev_nr[r + n];
}
}