#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_getRandomR(double* dev_mat, double* dev_ramR)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
extern __shared__ double matS[];

double *mat, *res;
mat = matS + threadIdx.x * 18;
res = mat  + 9;

mat[0] = 0; mat[4] = 0; mat[8] = 0;
mat[5] = dev_ramR[tid * 4 + 1];
mat[6] = dev_ramR[tid * 4 + 2];
mat[1] = dev_ramR[tid * 4 + 3];
mat[7] = -mat[5];
mat[2] = -mat[6];
mat[3] = -mat[1];

for(int i = 0; i < 9; i++)
res[i] = 0;

for (int i = 0; i < 3; i++)
for (int j = 0; j < 3; j++)
for (int k = 0; k < 3; k++)
res[i + j * 3] += mat[i + k * 3] * mat[k + j * 3];

double scale = 2 * dev_ramR[tid * 4];
for (int n = 0; n < 9; n++)
{
mat[n] *= scale;
mat[n] += res[n] * 2;
}

mat[0] += 1;
mat[4] += 1;
mat[8] += 1;

for (int n = 0; n < 9; n++)
{
dev_mat[tid * 9 + n] = mat[n];
}
}