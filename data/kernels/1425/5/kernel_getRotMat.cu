#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_getRotMat(double* devRotm, double* devnR, int nR)
{
extern __shared__ double matS[];
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid >= nR)
return;

double *mat, *res;
mat = matS + threadIdx.x * 18;
res = mat  + 9;

mat[0] = 0; mat[4] = 0; mat[8] = 0;
mat[5] = devnR[tid * 4 + 1];
mat[6] = devnR[tid * 4 + 2];
mat[1] = devnR[tid * 4 + 3];
mat[7] = -mat[5];
mat[2] = -mat[6];
mat[3] = -mat[1];

for(int i = 0; i < 9; i++)
res[i] = 0;

for (int i = 0; i < 3; i++)
for (int j = 0; j < 3; j++)
for (int k = 0; k < 3; k++)
res[i + j * 3] += mat[i + k * 3] * mat[k + j * 3];

double scale = 2 * devnR[tid * 4];
for (int n = 0; n < 9; n++)
{
mat[n] *= scale;
mat[n] += res[n] * 2;
}

mat[0] += 1;
mat[4] += 1;
mat[8] += 1;

for (int n = 0; n < 9; n++)
{
devRotm[tid * 9 + n] = mat[n];
}
}