#include "hip/hip_runtime.h"
#include "includes.h"
// System includes

// CUDA runtime

// Helper functions and utilities to work with CUDA


//template <int BLOCK_SIZE> __global__ void




uint32_t h_C[169] = { 0 };




__global__ void matrixMulCUDA(int *A, int *B, int *C)
{
//const int BLOCK_SIZE = 13;
// Block index
//int bx = blockIdx.x;
//int by = blockIdx.y;

// Thread index
int row = threadIdx.x;
int col = threadIdx.y;

int multi = 0;

for (int j = 0; j < 13; j++) {
multi += A[(row * 13) + j] * B[col + (13 * j)];
}
__syncthreads();
C[(row*13)+col] = multi + A[(row * 13) + col] + B[(row * 13)+col];
}