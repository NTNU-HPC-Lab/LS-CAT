#include "hip/hip_runtime.h"
#include "includes.h"


// Constants used by the program
#define BLOCK_DIM                      16


/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/




/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist        distance matrix
* @param ind         index matrix
* @param width       width of the distance matrix and of the index matrix
* @param height      height of the distance matrix and of the index matrix
* @param k           number of neighbors to consider
*/




/**
* K nearest neighbor algorithm
* - Initialize CUDA
* - Allocate device memory
* - Copy point sets (reference and query points) from host to device memory
* - Compute the distances + indexes to the k nearest neighbors for each query point
* - Copy distances from device to host memory
*
* @param ref_host      reference points ; pointer to linear matrix
* @param ref_nb        number of reference points ; width of the matrix
* @param query_host    query points ; pointer to linear matrix
* @param query_nb      number of query points ; width of the matrix
* @param dim           dimension of points ; height of the matrices
* @param k             number of neighbor to consider
* @param dist_host     distances to k nearest neighbors ; pointer to linear matrix
* @param dist_host     indexes of the k nearest neighbors ; pointer to linear matrix
*
*/
__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k){

// Variables
int l, i, j;
float *p_dist;
long  *p_ind;
float curr_dist, max_dist;
long  curr_row,  max_row;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){
// Pointer shift, initialization, and max value
p_dist   = dist + xIndex;
p_ind    = ind  + xIndex;
max_dist = p_dist[0];
p_ind[0] = 1;

// Part 1 : sort kth firt elementZ
for (l=1; l<k; l++){
curr_row  = l * width;
curr_dist = p_dist[curr_row];
if (curr_dist<max_dist){
i=l-1;
for (int a=0; a<l-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=l; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
} else {
p_ind[l*width] = l+1;
}
max_dist = p_dist[curr_row];
}

// Part 2 : insert element in the k-th first lines
max_row = (k-1)*width;
for (l=k; l<height; l++){
curr_dist = p_dist[l*width];
if (curr_dist<max_dist){
i=k-1;
for (int a=0; a<k-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=k-1; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
max_dist             = p_dist[max_row];
}
}
}
}