#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n*bsize)
trg[id] = src[idx[id/bsize]*bsize+id%bsize] * mult;
}