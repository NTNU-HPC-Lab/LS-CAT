#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaAcc_GPS_kernel_mod3( int NumDataPoints, float2* FreqData, float* PowerSpectrum)
{
const int sidx = (blockIdx.x * blockDim.x + threadIdx.x);

float ax,ay;

if ( sidx < NumDataPoints )
{
ax = FreqData[sidx].x;
ay = FreqData[sidx].y;
PowerSpectrum[sidx] =  __fadd_rn( __fmul_rn(ax,ax),__fmul_rn(ay,ay));
}
}