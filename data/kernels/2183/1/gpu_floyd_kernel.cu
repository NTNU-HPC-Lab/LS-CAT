#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_floyd_kernel(int k, int* adjacency_mtx, int* paths, int size)
{
int col = blockIdx.x * blockDim.x + threadIdx.x;
if (col >= size)return;
int idx = size * blockIdx.y + col;

__shared__ int best;
if (threadIdx.x == 0)
best = adjacency_mtx[size * blockIdx.y + k];
__syncthreads();
if (best == INF)
return;
int tmp_b = adjacency_mtx[k * size + col];
if (tmp_b == INF)
return;
int cur = best + tmp_b;
if (cur < adjacency_mtx[idx]) {
adjacency_mtx[idx] = cur;
paths[idx] = k;
}
}