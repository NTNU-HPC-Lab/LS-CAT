#include "hip/hip_runtime.h"
#include "includes.h"
//
// Created by Sowmya Parameshwara on 11/10/16.
//

/**
*
*  1) Input is stored by transposing the matrix, so that the attributes of a column are stored in a single row. This
*      will optimise the algorithm since all threads in a block will access nearby elements, while normalising.
*  2) Each row is normalised at a time for calculating standardscore, the calculated values are stored in output matrix by transposing.
*  3) Number of threads in a block is set as 16 (This value determined by checking performance for different values). The number of blocks
*     is decided based on matrix size "N" and number of threads.
*  4) The contents of a row are divided among the blocks. In each block,Each thread populates one elements of the block into shared data.
*     We then calculate partial sum without divergence, on the data stored in shared memory.
*  5) Once all blocks compute partial sum, we launch a kernel function on a single block by passing the calculated values from the previous step.
*     This will calculate the final sum and final squared sum. To this final block we ensure the size of the partial sum array passed equals
the next nearest power of 2 of "the number of blocks", as partial sum algorithm works only for powers of 2.
*  6)  The above data is used to calculate standard deviation for that row using the formula ((totalSquareSum + N*powf(mean, 2.0) - 2 * mean * totalSum)/(float)N)
*  7)  The above value is used to calculate standard score for every element in that row.
*  8)  The above step repeats for every row, calculating the standard score for all elements in the row.
*
*  Steps to compile and execute on Jarvis :
*  1)  qlogin -q interactive.q  (Launches interactive session).
*  2)  nvcc matrixNorm.cu -o matrixNorm (Compile code on jarvis).
*  3)  cd hw4 (Code is available here).
*  4) ./matrixNorm 15000 4   <Argument 1 : Size of matrix, Argument 2 : Random seed value>
*/


/* Program Parameters */
#define MAXN 15000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
__global__ void block_sum(const float *hostInput, float *sumResults, float *squareResults, const size_t n)
{
__shared__ float sharedSumData[1024];
__shared__ float sharedSquareData[1024];

int i = blockIdx.x * blockDim.x + threadIdx.x;
int tx = threadIdx.x;
float x = 0;
if(i < n) {
x = hostInput[i];
}
sharedSumData[tx] = x;
sharedSquareData[tx] = x*x;
__syncthreads();

// block-wide reduction in _shared_ mem
for(int offset = blockDim.x / 2;
offset > 0;
offset >>= 1)
{
if(tx < offset)
{
sharedSumData[tx] += sharedSumData[tx + offset];
sharedSquareData[tx] += sharedSquareData[tx + offset];
}
__syncthreads();
}

// finally, thread 0 writes the calculated result of this block
if(threadIdx.x == 0)
{
// note that the result is per-block
// not per-thread
sumResults[blockIdx.x] = sharedSumData[0];
squareResults[blockIdx.x] = sharedSquareData[0];
}
}