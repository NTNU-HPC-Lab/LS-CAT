#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel_2 (unsigned char *buffer, int img_w, int img_h, int *histo)
{
int id_x = blockIdx.x * blockDim.x + threadIdx.x ;
int id_y = blockIdx.y * blockDim.y + threadIdx.y ;

atomicAdd (&histo[buffer[id_y*img_w + id_x]] , 1 );
}