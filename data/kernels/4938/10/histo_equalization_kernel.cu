#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_equalization_kernel ( unsigned char *buffer, long size, int *histo, unsigned char *output ) {

int i = threadIdx.x + blockIdx.x * blockDim.x;
int offset = blockDim.x * gridDim.x;
while (i < size) {
if ( dev_lut[buffer[i]] > 255)
output[i] = 255;
else
output[i] = (unsigned char) dev_lut[buffer[i]];

i += offset;
}
}