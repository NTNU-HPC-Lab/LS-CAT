#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel ( unsigned char *buffer, long size, int *histo )
{
__shared__ int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x + blockIdx.x * blockDim.x;
int offset = blockDim.x * gridDim.x;
while (i < size)
{
atomicAdd( &temp[buffer[i]], 1);
i += offset;
}
__syncthreads();


atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}