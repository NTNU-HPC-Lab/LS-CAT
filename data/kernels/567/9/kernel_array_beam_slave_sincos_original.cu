#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_array_beam_slave_sincos_original(int N, float r1, float r2, float r3, float *x, float *y, float *z, float *sum, int blockDim_2) {
unsigned int n=threadIdx.x; //+blockDim.x*blockIdx.x;
__shared__ float tmpsum[1000]; /* assumed to be size 2*Nx1 */
if (n<N) {
float ss,cc;
sincosf((r1*__ldg(&x[n])+r2*__ldg(&y[n])+r3*__ldg(&z[n])),&ss,&cc);
tmpsum[2*n]=ss;
tmpsum[2*n+1]=cc;
}
__syncthreads();

// Build summation tree over elements, handling case where total threads is not a power of two.
int nTotalThreads = blockDim_2; // Total number of threads (==N), rounded up to the next power of two
while(nTotalThreads > 1) {
int halfPoint = (nTotalThreads >> 1); // divide by two
if (n < halfPoint) {
int thread2 = n + halfPoint;
if (thread2 < blockDim.x) { // Skipping the fictitious threads >N ( blockDim.x ... blockDim_2-1 )
tmpsum[2*n] = tmpsum[2*n]+tmpsum[2*thread2];
tmpsum[2*n+1] = tmpsum[2*n+1]+tmpsum[2*thread2+1];
}
}
__syncthreads();
nTotalThreads = halfPoint; // Reducing the binary tree size by two
}

/* now thread 0 will add up results */
if (threadIdx.x==0) {
sum[0]=tmpsum[0];
sum[1]=tmpsum[1];
}
}