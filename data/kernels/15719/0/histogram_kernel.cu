#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void histogram_kernel( float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist )
{
int _id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;

if (_id < c * size) {
int id = _id % size, dc = _id / size;

if (mask[id] < EPS)
return ;

float val  = I[_id];

float _minI = minI[dc];
float _maxI = maxI[dc];


if (_minI == _maxI) {
_minI -= 1;
_maxI += 1;
}

if (_minI <= val && val <= _maxI) {
int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins-1);
int index = dc * nbins + idx;
atomicAdd(&hist[index], 1.0f);
}

}

return ;
}