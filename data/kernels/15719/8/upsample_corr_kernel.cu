#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void upsample_corr_kernel( int *curr_corr, int *next_corr, int curr_h, int curr_w, int next_h, int next_w )
{
int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < next_h * next_w) {
int next_x = id % next_w, next_y = id / next_w;

float w_ratio = (float)next_w / (float)curr_w;
float h_ratio = (float)next_h / (float)curr_h;

int curr_x = (next_x + 0.5) / w_ratio;
int curr_y = (next_y + 0.5) / h_ratio;

curr_x = MAX(MIN(curr_x, curr_w-1), 0);
curr_y = MAX(MIN(curr_y, curr_h-1), 0);

int curr_id = curr_y * curr_w + curr_x;

int curr_x2 = curr_corr[2 * curr_id + 0];
int curr_y2 = curr_corr[2 * curr_id + 1];

int next_x2 = next_x + (curr_x2 - curr_x) * w_ratio + 0.5;
int next_y2 = next_y + (curr_y2 - curr_y) * h_ratio + 0.5;

next_x2 = MAX(MIN(next_x2, next_w-1), 0);
next_y2 = MAX(MIN(next_y2, next_h-1), 0);

next_corr[2 * id + 0] = next_x2;
next_corr[2 * id + 1] = next_y2;
}

return ;
}