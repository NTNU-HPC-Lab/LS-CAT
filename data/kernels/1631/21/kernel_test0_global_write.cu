#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{

unsigned int* ptr = (unsigned int*)_ptr;
unsigned int* end_ptr = (unsigned int*)_end_ptr;
unsigned int* orig_ptr = ptr;

unsigned int pattern = 1;

unsigned long mask = 4;

*ptr = pattern;

while(ptr < end_ptr){

ptr = (unsigned int*) ( ((unsigned long)orig_ptr) | mask);
if (ptr == orig_ptr){
mask = mask <<1;
continue;
}
if (ptr >= end_ptr){
break;
}

*ptr = pattern;

pattern = pattern << 1;
mask = mask << 1;
}
return;
}