#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{

unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);;
unsigned int* ptr = orig_ptr;
if (ptr >= (unsigned int*) end_ptr) {
return;
}

unsigned int* block_end = orig_ptr + BLOCKSIZE/sizeof(unsigned int);

unsigned int pattern = 1;

unsigned long mask = 4;

*ptr = pattern;

while(ptr < block_end){

ptr = (unsigned int*) ( ((unsigned long)orig_ptr) | mask);
if (ptr == orig_ptr){
mask = mask <<1;
continue;
}
if (ptr >= block_end){
break;
}

*ptr = pattern;

pattern = pattern << 1;
mask = mask << 1;
}
return;
}