#include "hip/hip_runtime.h"
#include "includes.h"
//
//  imgproc_main.cpp
//
//
//  Created by Nathaniel Lewis on 3/8/12.
//  Copyright (c) 2012 E1FTW Games. All rights reserved.
//



// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

// converts the pythagoran theorem along a vector on the GPU

// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

float af = float(a[idx]);
float bf = float(b[idx]);

c[idx] = (unsigned char) sqrtf(af*af + bf*bf);
}