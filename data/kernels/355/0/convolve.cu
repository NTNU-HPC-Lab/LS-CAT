#include "hip/hip_runtime.h"
#include "includes.h"
//
//  imgproc_main.cpp
//
//
//  Created by Nathaniel Lewis on 3/8/12.
//  Copyright (c) 2012 E1FTW Games. All rights reserved.
//



// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

// converts the pythagoran theorem along a vector on the GPU

// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, ssize_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
// Calculate our pixel's location
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

float sum = 0.0;
int   pWidth = kWidth/2;
int   pHeight = kHeight/2;

// Only execute for valid pixels
if(x >= pWidth+paddingX &&
y >= pHeight+paddingY &&
x < (blockDim.x * gridDim.x)-pWidth-paddingX &&
y < (blockDim.y * gridDim.y)-pHeight-paddingY)
{
for(int j = -pHeight; j <= pHeight; j++)
{
for(int i = -pWidth; i <= pWidth; i++)
{
// Sample the weight for this location
int ki = (i+pWidth);
int kj = (j+pHeight);
float w  = convolutionKernelStore[(kj * kWidth) + ki + kOffset];


sum += w * float(source[((y+j) * width) + (x+i)]);
}
}
}

// Average the sum
destination[(y * width) + x] = (unsigned char) sum;
}