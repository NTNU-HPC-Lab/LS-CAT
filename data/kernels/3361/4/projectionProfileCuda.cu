#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void projectionProfileCuda( const uint8_t * image, uint32_t rowSize, bool horizontal, uint32_t width, uint32_t height, uint32_t * projection )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
projection[image[y * rowSize + x]] = image[y * rowSize + x];
}
}