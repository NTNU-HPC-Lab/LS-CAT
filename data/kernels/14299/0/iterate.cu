#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADSPBLK 1024
#define THREADSPSM 2048
#define TILE_WIDTH 32
#define TOTAL_ITERATIONS 50

int main_n;


__global__ void iterate(float* originalMatrixD, float* solutionD, int originalMatrixWidth, int startingIndex) {
// __shared__ float originalMatrixDS [TILE_WIDTH][TILE_WIDTH];
__shared__ float originalMatrixDS [TILE_WIDTH * TILE_WIDTH];

int tx = threadIdx.x;
int ty = threadIdx.y;

int blockId = blockIdx.x + blockIdx.y * gridDim.x;

int currentMatrixIndex = blockId * (blockDim.x * blockDim.y) +
(threadIdx.y * blockDim.x) + threadIdx.x;

currentMatrixIndex += startingIndex;

originalMatrixDS[ty * TILE_WIDTH + tx] = originalMatrixD[currentMatrixIndex];

// Sync up w/ shared data set up
__syncthreads();

float replaceAmount;
bool onEdge = false;
int XEdgeCheckMod = currentMatrixIndex % originalMatrixWidth;

// X = 0 edge
if ( XEdgeCheckMod == 0) {
onEdge = true;
}

// X = N - 1
else if ( XEdgeCheckMod == (originalMatrixWidth - 1)) {
onEdge = true;
}

// Y = 0
else if (currentMatrixIndex < originalMatrixWidth) {
onEdge = true;
}

// Y = N - 1
else if (currentMatrixIndex >= (originalMatrixWidth * originalMatrixWidth
- originalMatrixWidth)) {
onEdge = true;
}

if (onEdge) {
replaceAmount = originalMatrixDS[ty * TILE_WIDTH + tx];
}

else {
// Top and Bottom come from Global memory
float top = originalMatrixD[currentMatrixIndex - originalMatrixWidth];
float bottom = originalMatrixD[currentMatrixIndex + originalMatrixWidth];
float left;
float right;

// Left and right edge come from Global memory
if (tx == 0 && ty == 0) {
left = originalMatrixD[currentMatrixIndex - 1];
}

else {
left = originalMatrixDS[ty * TILE_WIDTH + tx - 1];
}

if ((ty == TILE_WIDTH - 1) && (tx == TILE_WIDTH - 1)) {
right = originalMatrixD[currentMatrixIndex + 1];
}

else {
right = originalMatrixDS[ty * TILE_WIDTH + tx + 1];
}

replaceAmount = (left + right + top + bottom) / 4;
}

solutionD[currentMatrixIndex] = replaceAmount;
}