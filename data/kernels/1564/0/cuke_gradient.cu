#include "hip/hip_runtime.h"
#include "includes.h"



#define PI 3.1415926536f

texture<float, 2, hipReadModeElementType> texRef;
/*
* Paint a 2D texture with a moving red/green hatch pattern on a
* strobing blue background.  Note that this kernel reads to and
* writes from the texture, hence why this texture was not mapped
* as WriteDiscard.
*/

//=================================
// write to texture;
//=================================
enum colors
{
RED, GREEN, BLUE, ALPHA
};


__global__ void cuke_gradient(unsigned char *surface, int width, int height, size_t pitch, float t)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// in the case where, due to quantization into grids, we have
// more threads than pixels, skip the threads which don't
// correspond to valid pixels
if (x >= width || y >= height) return;

// get a pointer to the pixel at (x,y)
float* pixel = (float *)(surface + y*pitch) + 4*x;

pixel[RED]		= x/640.0f;
pixel[GREEN]	= y/480.0f;
pixel[BLUE]		= 0.0f;
pixel[ALPHA]	= 1.0f;
}