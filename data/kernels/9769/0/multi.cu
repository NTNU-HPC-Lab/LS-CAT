#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multi(float *a, float *b, float *c, int width) {
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

float result = 0;

if (col < width && row < width) {
for (int k = 0; k < width; k++) {
result += a[row * width + k] * b[k * width + col];
}
c[row * width + col] = result;
}
}