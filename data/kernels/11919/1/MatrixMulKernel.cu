#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
//2D Thread ID
int tx = threadIdx.x;
int ty = threadIdx.y;

//Pvalue stores the Pd element that is computed by the thread
float Pvalue = 0;

for(int k = 0; k < Width ; ++k) {
float Mdelement = Md[ty*Width + k];
float Ndelement = Nd[k*Width + tx];
Pvalue += (Mdelement*Ndelement);
}

Pd[ty*Width + tx] = Pvalue;
}