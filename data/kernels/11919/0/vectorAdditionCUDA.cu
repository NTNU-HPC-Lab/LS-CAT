#include "hip/hip_runtime.h"
#include "includes.h"

//no need for extern c according to stackoverflow answer by nvidia employee
extern "C"

__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
int ii = blockDim.x * blockIdx.x + threadIdx.x;
if (ii < n)
c[ii] = a[ii] + b[ii];
}