#include "hip/hip_runtime.h"
#include "includes.h"
/*
*  file name: mm_omp_vs_cuda.cu
*
*  mm_omp_vs_cuda.cu contains the code that realize some common used matrix operations in CUDA, and
*  an implementation of matrix multiplication speedup via openmp, this is a practice to compare the
*  of performance of cuda and openmp, as well as a trail of using cuda and openmp in the same program
*
*  this is a toy program for learning CUDA, some functions are reusable in other project
*  note:
*       compile: nvcc -Xcompiler \-fopenmp -lgomp mm_omp_vs_cuda.cu
*/
#define BLOCK_SIZE 16

/*
*********************************************************************
function name: gpu_matrix_mult

description: dot product of two matrix (not only square)

parameters:
&a GPU device pointer to a m X n matrix (A)
&b GPU device pointer to a n X k matrix (B)
&c GPU device output purpose pointer to a m X k matrix (C)
to store the result

Note:
grid and block should be configured as:
dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/

/*
*********************************************************************
function name: cpu_matrix_mult

description: dot product of two matrix (not only square) in CPU,
for validating GPU results

parameters:
&a CPU device pointer to a n X n matrix (A)
&b CPU device pointer to a n X n matrix (B)
&c CPU device output purpose pointer to a n X n matrix (C)
to store the result
Note:
grid and block should be configured as:

dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/

/*
*********************************************************************
function name: gpu_matrix_transpose

description: matrix transpose

parameters:
&mat_in GPU device pointer to a rows X cols matrix
&mat_out GPU device output purpose pointer to a cols X rows matrix
to store the result
Note:
grid and block should be configured as:
dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/
/*
*********************************************************************
function name: cpu_matrix_mult

description: dot product of two matrix (not only square) in CPU,
for validating GPU results

parameters:
&a CPU host pointer to a m X n matrix (A)
&b CPU host pointer to a n X k matrix (B)
&c CPU host output purpose pointer to a m X k matrix (C)
to store the result
return: none
*********************************************************************
*/
__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if( col < k && row < m)
{
for(int i = 0; i < n; i++)
{
sum += a[row * n + i] * b[i * k + col];
}
c[row * k + col] = sum;
}
}