#include "hip/hip_runtime.h"
#include "includes.h"
/*
*  file name: mm_omp_vs_cuda.cu
*
*  mm_omp_vs_cuda.cu contains the code that realize some common used matrix operations in CUDA, and
*  an implementation of matrix multiplication speedup via openmp, this is a practice to compare the
*  of performance of cuda and openmp, as well as a trail of using cuda and openmp in the same program
*
*  this is a toy program for learning CUDA, some functions are reusable in other project
*  note:
*       compile: nvcc -Xcompiler \-fopenmp -lgomp mm_omp_vs_cuda.cu
*/
#define BLOCK_SIZE 16

/*
*********************************************************************
function name: gpu_matrix_mult

description: dot product of two matrix (not only square)

parameters:
&a GPU device pointer to a m X n matrix (A)
&b GPU device pointer to a n X k matrix (B)
&c GPU device output purpose pointer to a m X k matrix (C)
to store the result

Note:
grid and block should be configured as:
dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/

/*
*********************************************************************
function name: cpu_matrix_mult

description: dot product of two matrix (not only square) in CPU,
for validating GPU results

parameters:
&a CPU device pointer to a n X n matrix (A)
&b CPU device pointer to a n X n matrix (B)
&c CPU device output purpose pointer to a n X n matrix (C)
to store the result
Note:
grid and block should be configured as:

dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/

/*
*********************************************************************
function name: gpu_matrix_transpose

description: matrix transpose

parameters:
&mat_in GPU device pointer to a rows X cols matrix
&mat_out GPU device output purpose pointer to a cols X rows matrix
to store the result
Note:
grid and block should be configured as:
dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

return: none
*********************************************************************
*/
/*
*********************************************************************
function name: cpu_matrix_mult

description: dot product of two matrix (not only square) in CPU,
for validating GPU results

parameters:
&a CPU host pointer to a m X n matrix (A)
&b CPU host pointer to a n X k matrix (B)
&c CPU host output purpose pointer to a m X k matrix (C)
to store the result
return: none
*********************************************************************
*/
__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n)
{
__shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
__shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
int tmp = 0;
int idx;

for (int sub = 0; sub < gridDim.x; ++sub)
{
idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
if(idx >= n*n)
{
// n may not divisible by BLOCK_SIZE
tile_a[threadIdx.y][threadIdx.x] = 0;
}
else
{
tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
}

idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
if(idx >= n*n)
{
tile_b[threadIdx.y][threadIdx.x] = 0;
}
else
{
tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
}
__syncthreads();

for (int k = 0; k < BLOCK_SIZE; ++k)
{
tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
}
__syncthreads();
}
if(row < n && col < n)
{
d_result[row * n + col] = tmp;
}
}