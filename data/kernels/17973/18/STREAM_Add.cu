#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Add(float *a, float *b, float *c,  size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
c[idx] = a[idx]+b[idx];
idx   += blockDim.x * gridDim.x;
}
}