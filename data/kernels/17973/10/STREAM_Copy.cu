#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Copy(float *a, float *b, size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
b[idx] = a[idx];
idx   += blockDim.x * gridDim.x;
}
}