#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad_Optimized_double(double *a, double *b, double *c, double scalar, size_t len)
{
/*
* Ensure size of thread index space is as large as or greater than
* vector index space else return.
*/
if (blockDim.x * gridDim.x < len) return;
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < len) c[idx] = a[idx]+scalar*b[idx];
}