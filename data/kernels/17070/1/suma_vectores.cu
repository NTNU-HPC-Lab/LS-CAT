#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void suma_vectores(int* c, const int* a, const int* b, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) {
c[i] = a[i] + b[i];
}
}