#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMult(int* m,int* n, int* p, int size)
{
//	matrixMult<<< grid_size,block_size >>>(d_m,d_n,d_p,n);

int row=blockIdx.y*blockDim.y+threadIdx.y;
int col=blockIdx.x*blockDim.x+threadIdx.x;
int p_sum;

for(int i=0;i<size;i++){
p_sum += m[row*size+i] * n[col*size+i];
}
p[row*size+col]=p_sum;
}