#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMult(int* m,int* n,int* p, int size)
{
int row=blockIdx.y*blockDim.y+threadIdx.y;
int col=blockIdx.x*blockDim.x+threadIdx.x;
int p_sum;

for(int i=0;i<size;i++){
p_sum += m[row*size+i] * n[col*size+i];

/*
si blockIdx.y= 0 entonces threadIdx.y se mueve::si blockIdx.x= 0 entonces threadIdx.x se mueve
*/
}
p[row*size+col]=p_sum;
}