#include "hip/hip_runtime.h"
#include "includes.h"



// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int* c, const int* a, const int* b, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) {
c[i] = a[i] + b[i];
}
}