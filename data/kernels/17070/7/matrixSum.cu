#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSum(int* a,int* b, int* c, int size)
{
// int max = maxThreadsPerBlock;
// printf("ERROR en global\n");
int pos = threadIdx.x + blockIdx.x * blockDim.x;
// printf("Block: %d\n", blockIdx.x );
// printf("pos= %d\n",pos);
if(pos<size*size){
c[pos] = a[pos] + b[pos];
}
}