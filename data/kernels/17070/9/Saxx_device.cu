#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Saxx_device(float* x, float* c, float xb, int n)
{
int i = threadIdx.x;
if (i < n)
c[i] = (x[i] - xb) * (x[i] - xb);

}