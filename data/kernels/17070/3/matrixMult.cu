#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMult(int* m, int* n, int* p, int size)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int p_sum;

for (int i = 0;i < size;i++) {
p_sum += m[row * size + i] * n[col * size + i];
}
p[row * size + col] = p_sum;
}