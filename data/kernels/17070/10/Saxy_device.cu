#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Saxy_device(float* x, float* y, float* d, float xb, float yb, int n)
{
int i = threadIdx.x;

if (i < n)
d[i] = (x[i] - xb) * (y[i] - yb);

}