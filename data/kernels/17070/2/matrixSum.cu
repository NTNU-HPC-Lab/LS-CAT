#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSum(int* a, int* b, int* c, int size)
{
// printf("ERROR en global\n");
int pos = threadIdx.x;
if (pos < size * size) {
c[pos] = a[pos] + b[pos];
}
}