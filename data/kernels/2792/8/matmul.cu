#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul(const float_t *A, const float_t *B, float_t *C, const float_t alpha, const float_t beta, int n, int d, int k, int max_block_rows) {
extern __shared__ __align__(sizeof(float_t)) unsigned char my_smem[];
float_t *shared = reinterpret_cast<float_t *>(my_smem);

float_t *s_A = shared;
float_t *s_B = shared + max_block_rows * d;

for (int i = threadIdx.x; i < d * k; i += blockDim.x) {
s_B[i] = B[i];
}

size_t block_start_row_index = blockIdx.x * max_block_rows;
size_t block_rows = max_block_rows;

if (blockIdx.x == gridDim.x - 1 && n % max_block_rows != 0) {
block_rows = n % max_block_rows;
}

for (size_t i = threadIdx.x; i < d * block_rows; i += blockDim.x) {
s_A[i] = alpha * A[d * block_start_row_index + i];
}

__syncthreads();

float_t elem_c = 0;

int col_c = threadIdx.x % k;
size_t abs_row_c = block_start_row_index + threadIdx.x / k;
int row_c = threadIdx.x / k;

// Thread/Block combination either too far for data array
// Or is calculating for index that should be calculated in a different
// blocks - in some edge cases "col_c * n + abs_row_c" can yield same
// result in different thread/block combinations
if (abs_row_c >= n || threadIdx.x >= block_rows * k) {
return;
}

for (size_t i = 0; i < d; i++) {
elem_c += s_B[d * col_c + i] * s_A[d * row_c + i];
}

C[col_c * n + abs_row_c] = beta * C[col_c * n + abs_row_c] + elem_c;
}