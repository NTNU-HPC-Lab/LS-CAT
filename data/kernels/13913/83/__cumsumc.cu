#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __cumsumc(int nrows, int ncols, double *A, double *B) {
__shared__ double buff[32];
int i, j, k, lim;
double v, sum;
int icol = threadIdx.y + blockDim.y * blockIdx.x;
__syncthreads();
for (i = icol; i < ncols; i += blockDim.y * gridDim.x) {
sum = 0.0f;
for (j = 0; j < nrows; j += blockDim.x) {
v = 0;
if (j + threadIdx.x < nrows) {
v = A[j + threadIdx.x + i * nrows];
}
__syncthreads();
buff[threadIdx.x] = v;
lim = min(blockDim.x, nrows - j);
#pragma unroll
for (k = 1; k < lim; k = k + k) {
__syncthreads();
if (threadIdx.x >= k) {
v += buff[threadIdx.x - k];
}
__syncthreads();
buff[threadIdx.x] = v;
}
v += sum;
if (j + threadIdx.x < nrows) {
B[j + threadIdx.x + i * nrows] = v;
}
__syncthreads();
sum = buff[31];
__syncthreads();
}
}
}