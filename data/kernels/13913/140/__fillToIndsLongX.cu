#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToIndsLongX(long long A, long long *B, long long len) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
long long i;
for (i = tid; i < len; i += step) {
B[i] = A;
}
}