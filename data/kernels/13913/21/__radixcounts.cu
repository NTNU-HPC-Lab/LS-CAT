#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __radixcounts(float *a, int n, int digit, unsigned int *bi) {
__shared__ unsigned int ic[RNDVALS];

int istart = (int)(((long long)blockIdx.x) * n / gridDim.x);
int iend = (int)(((long long)(blockIdx.x+1)) * n / gridDim.x);
int tid = threadIdx.x;
int bibase = RNDVALS * (blockIdx.x + istart / RBIGBLK);

for (int i = istart; i < iend; i += RBIGBLK) {

__syncthreads();
ic[threadIdx.x] = 0;
__syncthreads();
for (int j = i + tid; j < min(iend, i+tid+RBIGBLK); j += RNTHREADS) {
float v = a[j];
unsigned char *cv = (unsigned char *)&v;
atomicInc(&ic[cv[digit]], 65536*32767);
}
__syncthreads();
bi[bibase + threadIdx.x] = ic[threadIdx.x];
bibase += RNDVALS;
}
}