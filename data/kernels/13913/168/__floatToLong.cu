#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __floatToLong(float *A, long long *B, int N) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
B[i] = (float)(A[i]);
}
}