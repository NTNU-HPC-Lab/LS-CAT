#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __embedmat2d(float *a, long long *b, int nrows, int ncols, int sortdown) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
const int signbit = 0x80000000;
const int mag =     0x7fffffff;
int icol = 0;
for (int i = tid; i < nrows*ncols; i += blockDim.x*gridDim.x*gridDim.y) {
float v = a[i];
int vi = *((int *)&v);
if (vi & signbit) {
vi = -(vi & mag);
}
icol = i/nrows+1;
if (sortdown) icol = ncols - icol + 1;
b[i] = (long long)vi + (((long long)icol)<<32);
}
}