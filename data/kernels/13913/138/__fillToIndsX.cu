#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToIndsX(float A, float *B, long long len) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
long long i;
for (i = tid; i < len; i += step) {
B[i] = A;
}
}