#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __intToFloat(int *A, float *B, int N) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
B[i] = (float)(A[i]);
}
}