#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __full(int *ir, int *ic, double *data, double *od, int nrows, int ncols, int nnz) {
int i, row, col;
double v;
int id = threadIdx.x + blockIdx.x * blockDim.x;
for (i = id; i < nnz; i += blockDim.x * gridDim.x) {
v = data[i];
row = ir[i];
col = ic[i];
od[row + col * nrows] = v;
}
}