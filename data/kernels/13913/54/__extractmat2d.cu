#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __extractmat2d(double *a, long long *b, int nrows, int ncols) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
const int signbit = 0x80000000;
const int mag =     0x7fffffff;
for (int i = tid; i < nrows*ncols; i += blockDim.x*gridDim.x*gridDim.y) {
int vi = *((int *)&b[i]);
if (vi & signbit) {
vi = -(vi & mag);
}
a[i] = *((double *)&vi);
}
}