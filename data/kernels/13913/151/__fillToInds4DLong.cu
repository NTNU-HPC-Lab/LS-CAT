#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToInds4DLong(long long A, long long *B, int ldb, int rdb, int tdb, int *I, int nrows, int *J, int ncols, int *K, int nk, int *L, int nl) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
int l = tid / (nrows * ncols * nk);
int tidrem = tid - l * (nrows * ncols * nk);
int lstep = step / (nrows * ncols * nk);
int steprem = step - lstep * (nrows * ncols * nk);
int k = tidrem / (nrows * ncols);
tidrem = tidrem - k * (nrows * ncols);
int kstep = steprem / (nrows * ncols);
steprem = steprem - kstep * (nrows * ncols);
int j = tidrem / nrows;
int i = tidrem - j * nrows;
int jstep = steprem / nrows;
int istep = steprem - jstep * nrows;
int id, mapi, mapj, mapk, mapl;
for (id = tid; id < nrows * ncols * nk * nl; id += step) {
mapl = l;
if (L != NULL) mapl = L[l];
mapk = k;
if (K != NULL) mapk = K[k];
mapj = j;
if (J != NULL) mapj = J[j];
mapi = i;
if (I != NULL) mapi = I[i];
B[mapi + ldb * (mapj + rdb * (mapk + tdb * mapl))] = A;
i += istep;
if (i >= nrows) {i -= nrows; j++;}
j += jstep;
if (j >= ncols) {j -= ncols; k++;}
k += kstep;
if (k >= nk) {k -= nk; l++;}
l += lstep;
}
}