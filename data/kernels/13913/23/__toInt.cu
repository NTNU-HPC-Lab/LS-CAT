#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __toInt(double *A, int *B, int N) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
B[i] = (int)(A[i]);
}
}