#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToInds3D(double A, double *B, int ldb, int rdb, int *I, int nrows, int *J, int ncols, int *K, int nk) {
int ii = threadIdx.x + blockDim.x * blockIdx.x;
int jj = threadIdx.y + blockDim.y * blockIdx.y;
int kk = threadIdx.z + blockDim.z * blockIdx.z;
int i, j, k, mapi, mapj, mapk;
for (k = kk; k < nk; k += blockDim.z * gridDim.z) {
mapk = k;
if (K != NULL) mapk = K[k];
for (j = jj; j < ncols; j += blockDim.y * gridDim.y) {
mapj = j;
if (J != NULL) mapj = J[j];
if (I != NULL) {
for (i = ii; i < nrows; i += blockDim.x * gridDim.x) {
mapi = I[i];
B[mapi + ldb * (mapj + rdb * mapk)] = A;
}
} else {
for (i = ii; i < nrows; i += blockDim.x * gridDim.x) {
mapi = i;
B[mapi + ldb * (mapj + rdb * mapk)] = A;
}
}
}
}
}