#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToInds4D(double A, double *B, int ldb, int rdb, int tdb, int *I, int nrows, int *J, int ncols, int *K, int nk, int *L, int nl, int ntk, int nbk, int ntl, int nbl) {
int ii = threadIdx.x + blockDim.x * blockIdx.x;
int jj = threadIdx.y + blockDim.y * blockIdx.y;
int tk = threadIdx.z / ntl;
int tl = threadIdx.z - tk * ntl;
int bk = blockIdx.z / nbl;
int bl = blockIdx.z - bk * nbl;
int kk = tk + ntk * bk;
int ll = tl + ntl * bl;
int i, j, k, l, mapi, mapj, mapk, mapl;
for (l = ll; l < nl; l += ntl * nbl) {
mapl = l;
if (L != NULL) mapl = L[l];
for (k = kk; k < nk; k += ntk * nbk) {
mapk = k;
if (K != NULL) mapk = K[k];
for (j = jj; j < ncols; j += blockDim.y * gridDim.y) {
mapj = j;
if (J != NULL) mapj = J[j];
if (I != NULL) {
for (i = ii; i < nrows; i += blockDim.x * gridDim.x) {
mapi = I[i];
B[mapi + ldb * (mapj + rdb * (mapk + tdb * mapl))] = A;
}
} else {
for (i = ii; i < nrows; i += blockDim.x * gridDim.x) {
B[i + ldb * (mapj + rdb * (mapk + tdb * mapl))] = A;
}
}
}
}
}
}