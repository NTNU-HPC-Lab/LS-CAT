#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __floatToDouble(float *A, double *B, int N) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
B[i] = (double)(A[i]);
}
}