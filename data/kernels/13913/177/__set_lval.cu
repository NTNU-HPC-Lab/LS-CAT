#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __set_lval(long long *A, long long val, int length) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < length; i += blockDim.x * gridDim.x * gridDim.y) {
A[i] = val;
}
}