#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __embedmat(float *a, int *b, long long *c, int n) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
const int signbit = 0x80000000;
const int mag =     0x7fffffff;
for (int i = tid; i < n; i += blockDim.x*gridDim.x*gridDim.y) {
float v = a[i];
int vi = *((int *)&v);
if (vi & signbit) {
vi = -(vi & mag);
}
c[i] = (long long)vi + (((long long)b[i])<<32);
}
}