#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __extractmat(double *a, int *b, long long *c, int n) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
const int signbit = 0x80000000;
const int mag =     0x7fffffff;
for (int i = tid; i < n; i += blockDim.x*gridDim.x*gridDim.y) {
int vi = *((int *)&c[i]);
if (vi & signbit) {
vi = -(vi & mag);
}
a[i] = *((double *)&vi);
b[i] = *(((int *)&c[i])+1);
}
}