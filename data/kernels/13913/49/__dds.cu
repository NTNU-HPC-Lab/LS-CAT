#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __dds(int nrows, int nnz, double *A, double *B, int *Cir, int *Cic, double *P) {
__shared__ double parts[32*DDS_BLKY];
int jstart = ((long long)blockIdx.x) * nnz / gridDim.x;
int jend = ((long long)(blockIdx.x + 1)) * nnz / gridDim.x;
int tid = threadIdx.x + blockDim.x * threadIdx.y;
for (int j = jstart; j < jend ; j++) {
double sum = 0;
int aoff = nrows * Cir[j];
int boff = nrows * Cic[j];
for (int i = tid; i < nrows; i += blockDim.x * blockDim.y) {
sum += A[i + aoff] * B[i + boff];
}
parts[tid] = sum;
for (int i = 1; i < blockDim.x * blockDim.y; i *= 2) {
__syncthreads();
if (i + tid < blockDim.x * blockDim.y) {
parts[tid] = parts[tid] + parts[i + tid];
}
}
__syncthreads();
if (tid == 0) {
P[j] = parts[0];
}
__syncthreads();
}
}