#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __initSeq(int *A, int nrows, int ncols) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
A[i] = i % nrows;
}
}