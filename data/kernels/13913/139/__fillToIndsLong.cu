#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToIndsLong(long long A, long long *B, int *I, long long len) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
long long i;
for (i = tid; i < len; i += step) {
B[I[i]] = A;
}
}