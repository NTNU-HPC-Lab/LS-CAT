#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __transpose(double *in, int instride, double *out, int outstride, int nrows, int ncols) {
int nx = BLOCKDIM * gridDim.x;
int ny = BLOCKDIM * gridDim.y;
int ix = BLOCKDIM * blockIdx.x;
int iy = BLOCKDIM * blockIdx.y;
__shared__ double tile[BLOCKDIM][BLOCKDIM+1];

for (int yb = iy; yb < ncols; yb += ny) {
for (int xb = ix; xb < nrows; xb += nx) {
if (xb + threadIdx.x < nrows) {
int ylim = min(ncols, yb + BLOCKDIM);
for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
tile[threadIdx.x][y-yb] = in[threadIdx.x+xb + y*instride];
}
}
__syncthreads();
if (yb + threadIdx.x < ncols) {
int xlim = min(nrows, xb + BLOCKDIM);
for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
out[threadIdx.x + yb + x*outstride] = tile[x-xb][threadIdx.x];
}
}
__syncthreads();
}
}
}