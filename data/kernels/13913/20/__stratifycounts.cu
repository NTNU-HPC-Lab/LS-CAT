#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __stratifycounts(float *strata, int n,  float *a, unsigned int *bi) {
__shared__ unsigned int ic[SNDVALS][SNDGRPS];
__shared__ float ss[SNDVALS];
int istart = (int)(((long long)blockIdx.x) * n / gridDim.x);
int iend = (int)(((long long)(blockIdx.x+1)) * n / gridDim.x);
int bibase = SNDVALS * (blockIdx.x + istart / SBIGBLK);
int tid = threadIdx.x + threadIdx.y * blockDim.x;

if (threadIdx.y == 0) {
ss[threadIdx.x] = strata[threadIdx.x];
}
for (int i = istart; i < iend; i += SBIGBLK) {
__syncthreads();
if (threadIdx.y < SNDGRPS) {
ic[threadIdx.x][threadIdx.y] = 0;
}
__syncthreads();
for (int k = i + tid; k < min(iend, i + tid + SBIGBLK); k += SNTHREADS) {
float v = a[k];
int j = 0;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = (v > ss[j]) ? 2*j+2 : 2*j+1;
j = j - SNDVALS + 1;
atomicInc(&ic[j][threadIdx.y], 65536*32767);
}
__syncthreads();
if (threadIdx.y == 0) {
bi[bibase + threadIdx.x] = ic[threadIdx.x][0] + ic[threadIdx.x][1] + ic[threadIdx.x][2] + ic[threadIdx.x][3];
}
bibase += SNDVALS;
}
}