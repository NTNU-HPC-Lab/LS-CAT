#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToInds(double A, double *B, int *I, long long len) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
long long i;
for (i = tid; i < len; i += step) {
B[I[i]] = A;
}
}