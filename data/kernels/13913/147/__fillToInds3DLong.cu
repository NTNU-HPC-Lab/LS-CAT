#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __fillToInds3DLong(long long A, long long *B, int ldb, int rdb, int *I, int nrows, int *J, int ncols, int *K, int nk) {
int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
int step = blockDim.x * gridDim.x * gridDim.y;
int k = tid / (nrows * ncols);
int tidrem = tid - k * (nrows * ncols);
int kstep = step / (nrows * ncols);
int steprem = step - kstep * (nrows * ncols);
int j = tidrem / nrows;
int i = tidrem - j * nrows;
int jstep = steprem / nrows;
int istep = steprem - jstep * nrows;
int id, mapi, mapj, mapk;
for (id = tid; id < nrows * ncols * nk; id += step) {
mapk = k;
if (K != NULL) mapk = K[k];
mapj = j;
if (J != NULL) mapj = J[j];
mapi = i;
if (I != NULL) mapi = I[i];
B[mapi + ldb * (mapj + rdb * mapk)] = A;
i += istep;
if (i >= nrows) {i -= nrows; j++;}
j += jstep;
if (j >= ncols) {j -= ncols; k++;}
k += kstep;
}
}