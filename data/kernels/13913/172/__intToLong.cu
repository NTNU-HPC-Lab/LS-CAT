#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __intToLong(int *A, long long *B, int N) {
int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
B[i] = (long long)(A[i]);
}
}