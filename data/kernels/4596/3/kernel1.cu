#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel1(int* D, int* q, int b){

int i = threadIdx.x + b * THR_PER_BL;
int j = threadIdx.y + b * THR_PER_BL;

float d, f, e;
for(int k = b * THR_PER_BL; k < (b + 1) * THR_PER_BL; k++)
{
d = D[i * N + j];
f = D[i * N + k];
e = D[k * N + j];

__syncthreads();

if(d > f + e)
{
D[i * N + j] = f + e;
q[i * N + j] = k;
}
}
}