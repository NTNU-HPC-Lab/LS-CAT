#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel3(int* D, int* q, int b){

int i, j;

if(blockIdx.x >= b)
{
i = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
}
else
{
i = blockIdx.x * blockDim.x + threadIdx.x;
}
if(blockIdx.y >= b)
{
j = (blockIdx.y + 1) * blockDim.y + threadIdx.y;
}
else
{
j = blockIdx.y * blockDim.y + threadIdx.y;
}

float d, f, e;
for(int k = b * THR_PER_BL; k < (b + 1) * THR_PER_BL; k++)
{
d = D[i * N + j];
f = D[i * N + k];
e = D[k * N + j];

__syncthreads();

if(d > f + e)
{
D[i * N + j] = f + e;
q[i * N + j] = k;
}
}
}