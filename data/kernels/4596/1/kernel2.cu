#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright 2019, Dimitra S. Kaitalidou, All rights reserved


#define N 256
#define THR_PER_BL 8
#define BL_PER_GR 32



__global__ void kernel2(int* D, int* Q){

int i = blockIdx.x * blockDim.x + threadIdx.x;

// Assign the values of the output array back to the input array
D[i] = Q[i];
}