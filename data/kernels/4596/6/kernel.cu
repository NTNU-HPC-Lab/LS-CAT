#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* D, int* q, int k){

// Find index of i row and j column of the distance array
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

if(D[i * N + j] > D[i * N + k] + D[k * N + j])
{
D[i * N + j] = D[i * N + k] + D[k * N + j];
q[i * N + j] = k;
}
}