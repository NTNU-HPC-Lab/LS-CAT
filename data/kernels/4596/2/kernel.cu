#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* D, int* Q, int bits){

// Find index
int i = blockIdx.x * blockDim.x + threadIdx.x;

// Initialize variables that will be shifted left and right
int shifted_right = i;
int shifted_left = shifted_right;

// Perform bit reversal permutation
for(int a = 1; a < bits; a++)
{
shifted_right >>= 1;
shifted_left <<= 1;
shifted_left |= shifted_right & 1;
}
shifted_left &= N - 1;

// Assign the values to the bit reversed positions
Q[shifted_left] = D[i];
}