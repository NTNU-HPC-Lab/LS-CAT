#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2(int* D, int* q, int b){

int i, j;
if(blockIdx.y == 0)
{
j = b * blockDim.y + threadIdx.y;
if(blockIdx.x >= b)
{
i = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
}
else
{
i = blockIdx.x * blockDim.x + threadIdx.x;
}
}
else
{
i = b * blockDim.y + threadIdx.y;
if(blockIdx.x >= b)
{
j = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
}
else
{
j = blockIdx.x * blockDim.x + threadIdx.x;
}
}

float d, f, e;
for(int k = b * THR_PER_BL; k < (b + 1) * THR_PER_BL; k++)
{
d = D[i * N + j];
f = D[i * N + k];
e = D[k * N + j];

__syncthreads();

if(d > f + e)
{
D[i * N + j] = f + e;
q[i * N + j] = k;
}
}
}