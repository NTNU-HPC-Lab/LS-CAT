#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright 2019, Dimitra S. Kaitalidou, All rights reserved


#define N 256
#define THR_PER_BL 8
#define BL_PER_GR 32



__global__ void kernel1(int* D, int* Q, int k){

// Find index
int i = blockIdx.x * blockDim.x + threadIdx.x;
int block = (int)(i / (2 * k));
int j;

if(i % 2 == 0) j = 2 * block * k + (int)(i / 2) - k * ((int)(i / (2 * k)));
else j = (2 * block + 1) * k + (int)(i / 2) - k * ((int)(i / (2 * k)));

// Assign the values to the output array
Q[j] = D[i];
}