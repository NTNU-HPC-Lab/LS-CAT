#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SigmoidProbPolynomProbsImpl( const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs) {
if (threadIdx.x < batchSize) {
int polynomId = blockIdx.x;

features +=  threadIdx.x;
probs += threadIdx.x;

while (polynomId < polynomCount) {
int offset = polynomOffsets[polynomId];
int nextOffset = polynomOffsets[polynomId + 1];
const int depth = nextOffset - offset;

//            bool isTrue = true;
float logProb = 0;
for (int i = 0; i < depth; ++i) {
const int f = __ldg(splits + offset + i);
const float c = __ldg(conditions + offset + i);
const float x = __ldg(features + f * batchSize);
const float val = -lambda * (x - c);
//                isTrue = x <= c? false : isTrue;
const float expVal = 1.0f + expf(val);

//            p( split = 1) = 1.0 / (1.0 + exp(-(x - c)))
//            c = 0, x= inf, p = 1.0 / (1.0 + exp(-inf) = 0
//            log(p) = -log(1.0 + exp(-(x - c))
const float isTrueLogProb = isfinite(expVal) ? log(expVal) : val;
logProb -= isTrueLogProb;
}
const float prob = expf(logProb);
//            const float prob = isTrue ? 1 : 0;//exp(logProb);
probs[polynomId * batchSize] = prob;
polynomId += gridDim.x;
}
}
}