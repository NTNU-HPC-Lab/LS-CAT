#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExpProbPolynomProbsImpl( const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs) {
if (threadIdx.x < batchSize) {
int polynomId = blockIdx.x;

features +=  threadIdx.x;
probs += threadIdx.x;

while (polynomId < polynomCount) {
int offset = polynomOffsets[polynomId];
int nextOffset = polynomOffsets[polynomId + 1];
const int depth = nextOffset - offset;

float logProb = 0;
bool zeroProb = false;
for (int i = 0; i < depth; ++i) {
if (zeroProb) {
continue;
}

const int f = __ldg(splits + offset + i);
const float c = __ldg(conditions + offset + i);
const float x = __ldg(features + f * batchSize);

const float val = -lambda * x;
const float expVal = 1.0f - expf(val);

if (isfinite(log(expVal))) {
logProb += log(expVal);
} else {
zeroProb = true;
}
}

float prob = 0.0f;
if (!zeroProb) {
prob = expf(logProb);
}

probs[polynomId * batchSize] = prob;
polynomId += gridDim.x;
}
}
}