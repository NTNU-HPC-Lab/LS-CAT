#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PolynomForwardImpl( const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {

//out: batch_elem0 dim0, dim1, dimk batch_elem1 dim0 dim1 dimk
//so threads
int polynomId = blockIdx.x;
const int dimId = blockIdx.y;

int tid = threadIdx.x;
if (tid >= batchSize) {
return;
}

float sum = 0;
probs += threadIdx.x;
values += dimId;

while (polynomId < polynomCount) {
const float polynomProb = __ldg(probs + polynomId * batchSize);
const float out = __ldg(values + polynomId * outputDim);
sum += polynomProb * out;
polynomId += gridDim.x;
}

atomicAdd(out + dimId * batchSize + threadIdx.x, sum);
}