#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void conv(const float* gm, float* convolved, int bh, int bw, int ih, int iw, int ch, int cw, int smH, int smW, int k, float* sm, int gID, int tID, int nT, int rel_row, int rel_col, int nRows, int stopPrefetchRowID, int lastActiveThreadID) {

for(int i=k; i<=nRows; i++)
{
/*
----prefetch a pixel value from GM and store it in register----

all threads fetch the cell value immediately below to the current cell iteratively

last thread in the block would fetch k cells immediately below the current cell

boundary check would be needed for the blocks that act on the bottom most partition of the input image to prevent it from prefetching out of image values.
*/
float reg;
float regArr[K];
if(i <= stopPrefetchRowID){
reg = gm[i * iw + gID];
if(tID == lastActiveThreadID){
for(int j=1; j<=k-1; j++){
regArr[j] = gm[(i * iw) + gID + j];
}
}
}
// load k * k pixels above the current cell
float imgPixels[K*K];
for(int r=i-k; r<i; r++){
for(int c=0; c<k; c++){
/* translate the indices to [0,k] using r - (i-k) as imgPixels is of size k*k */
imgPixels[(r-i+k)*k + c] = sm[r * smW + tID + c];
}
}
/*multiply image pixel values with filter values (direct convolution) */
float convolvedCell = 0.0;
for(int c=0; c<k*k; c++){
convolvedCell += cm[c]*imgPixels[c];
}
//place the convolvedCell value into convolvedMatrix
int cID = ( ( (rel_row * bh) + (i-k) ) * cw )+( rel_col * nT )+tID;
if(cID < 0 || cID >= ch*cw ) {
printf("cID : %d, tID : %d, gID : %d\n", cID, tID, gID );
}
convolved[cID] = convolvedCell;
__syncthreads();
if(i <= stopPrefetchRowID){
sm[i * smW + tID] = reg;
if(tID == lastActiveThreadID){
for(int j=1; j<=k-1; j++){
int sID = i *smW + tID + j;
sm[sID] = regArr[j];
}
}
}
__syncthreads();
}


}
__global__ void conv_kernel(const float* gm, float* convolved, int bh, int bw, int ih, int iw, int ch, int cw, int smH, int smW, int k) {

int tID = threadIdx.x;
int bID = blockIdx.x;
int nT = blockDim.x;
int nB = gridDim.x;
int nBx = iw / nT;
//printf("num of blocks is %d\n", nB);
//printf("nB in a row is %d\n", nBx);
//check for right border or bottom border thread block
bool isBottomBorder = false;
bool isRightBorder = false;
// bottom border thread block
if(bID >= nB - nBx) {
//printf("bID : %d is bottom border\n", bID);
isBottomBorder = true;
}
// right border thread block
if((bID+1) % nBx == 0){
//printf("bID : %d is right border\n", bID);
isRightBorder = true;
}

// ---------------- Load k rows from GM into SM ----------------------

__shared__ float sm[ (BLCH + K - 1) * (BLCW + K - 1) ];
// rel_row and rel_col maps the Thread Block to appropriate position
int rel_row = bID / nBx;
int rel_col = bID % nBx;
// (rel_row * bh * iw) covers all the cells before row_ids bh, 2bh, 3bh ..
// gID finally maps threads to cells at rows 0, bh, 2bh, 3bh, ...
int gID = (rel_row * bh * iw) + (rel_col * nT) + tID;

for(int i=0; i<k; i++){

int sID = i * smW + tID;
sm[sID] = gm[i * iw + gID];
/* if last thread in the block, it should fetch additional k-1 pixels
in each row which are needed for computation of the convolution
*/
if(!isRightBorder && tID == nT-1){
for(int j=1; j<=k-1; j++){
sID = (i * smW) + tID + j;
sm[sID] = gm[i * iw + gID + j];
}
}

}

__syncthreads();

if( !isBottomBorder && !isRightBorder ){
int lastActiveThreadID = nT - 1;
int nRows = bh + k - 1;
int stopPrefetchRowID = nRows;
conv( gm, convolved, bh, bw,
ih, iw, ch, cw, smH, smW, k,
sm, gID, tID, nT, rel_row, rel_col,
nRows, stopPrefetchRowID, lastActiveThreadID );
}
else if( isBottomBorder && isRightBorder ){
/* make the last k-1 threads in the block to be idle. as there is no convolution needed for them */
if(tID < (nT - (k-1))){
int nRows = bh;
int stopPrefetchRowID = nRows - 1;
int lastActiveThreadID = nT - k;
conv( gm, convolved, bh, bw,
ih, iw, ch, cw, smH, smW, k,
sm, gID, tID, nT, rel_row, rel_col,
nRows, stopPrefetchRowID, lastActiveThreadID );
}
}
else if( isBottomBorder ){
int nRows = bh;
int stopPrefetchRowID = nRows-1;
int lastActiveThreadID = nT - 1;
conv( gm, convolved, bh, bw,
ih, iw, ch, cw, smH, smW, k,
sm, gID, tID, nT, rel_row, rel_col,
nRows, stopPrefetchRowID, lastActiveThreadID );


}
else if( isRightBorder ){
/* make the last k-1 threads in the block to be idle. as there is no convolution needed for them */
if(tID < (nT - (k-1))){
int nRows = bh + k - 1;
int stopPrefetchRowID = nRows;
int lastActiveThreadID = nT - k;
conv( gm, convolved, bh, bw,
ih, iw, ch, cw, smH, smW, k,
sm, gID, tID, nT, rel_row, rel_col,
nRows, stopPrefetchRowID, lastActiveThreadID );
}

}



}