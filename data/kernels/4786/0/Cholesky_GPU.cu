#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

const int MAX = 100;



__global__ void Cholesky_GPU(double *matrix, int n){

//n threads running in parallel

//int x = blockIdx.x;
int y = threadIdx.x;
//int i = x;
int j = y;

extern __device__ __shared__ double localMatrix[];
//	extern __device__ __shared__ double sum[];
//matrix2d[x][y] = matrix1d[x*n+y]

//Copy to shared mem

for(int i=0; i<n; i++)
localMatrix[i*n+j] = matrix[i*n+j];

localMatrix[n*n+j] = 0; // sum column
__syncthreads();

//Do the calc;
#pragma unroll
for(int i=0; i<n; i++){
if(j<i){
localMatrix[i*n+j] = 0;
}
if(j>=i) {
localMatrix[n*n+j]=0;//initialize sum to 0
for(int k=0; k<i; k++)
localMatrix[n*n+j] +=localMatrix[k*n+i]*localMatrix[k*n+j]; // sums
//if(j<i){
//	localMatrix[i*n+j]=0;
//}
if(i == j){
localMatrix[i*n+j] = sqrt(localMatrix[i*n+j] - localMatrix[n*n+j]);
}if(j > i){
localMatrix[i*n+j] = (localMatrix[i*n+j] - localMatrix[n*n+j])/localMatrix[i*n+i];
}
}
}


__syncthreads();


for(int i=0; i<n; i++)
matrix[i*n+j] = localMatrix[i*n+j];
//Copy back



}