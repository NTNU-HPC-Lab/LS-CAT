#include "hip/hip_runtime.h"
#include "includes.h"
//==========================================================================================================
// A small snippet of code to solve equation of types Ax=B using Gaussian Elimniation
// Author - Anmol Gupta, Naved Ansari
// Course - EC513 - Introduction to Computer Architecture
// Boston University
//==========================================================================================================

//==========================================================================================================
// Command to compile the code
//nvcc -o GaussianElimination GaussianElimination.cu
//==========================================================================================================


// Assertion to check for errors
__global__ void gauss_elimination_cuda(float *a_d, float *b_d ,int size) {
int idx = threadIdx.x;
int idy = threadIdx.y;

__shared__ float temp[40][40];
temp[idy][idx] = a_d[(idy * (size+1)) + idx];
__syncthreads();
//cuPrintf("T idy=%d, idx=%d, temp=%f\n", idy, idx, a_d[(idy * (size+1)) + idx]);

for(int column = 0; column < size-1; column++){
if(idy > column && idx >= column){
float t = temp[column][idx] - (temp[column][column] / temp[idy][column]) * temp[idy][idx];
__syncthreads();
temp[idy][idx] = t;
}
__syncthreads();
}

b_d[idy*(size+1) + idx] = temp[idy][idx];
}