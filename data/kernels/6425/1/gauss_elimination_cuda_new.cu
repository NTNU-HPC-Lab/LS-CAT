#include "hip/hip_runtime.h"
#include "includes.h"
//==========================================================================================================
// A small snippet of code to solve equation of types Ax=B using Gaussian Elimniation
// Author - Anmol Gupta, Naved Ansari
// Course - EC513 - Introduction to Computer Architecture
// Boston University
//==========================================================================================================

//==========================================================================================================
// Command to compile the code
//nvcc -o GaussianElimination GaussianElimination.cu
//==========================================================================================================


// Assertion to check for errors
__global__ void gauss_elimination_cuda_new(float *a_d, float *b_d ,int size) {
int i, j;
int idy = threadIdx.x;

__shared__ float temp[MAXSIZE+10][MAXSIZE+10];
//copy to share
for(i=0; i<size+1; i++){
temp[idy][i] = a_d[(idy * (size+1)) + i];
//cuPrintf("T idy=%d, num = %d, temp=%f\n", idy, i, temp[idy][i]);
}
__syncthreads();


//loop through every row, calculate every column in parallel
for(i=1; i<size; i++){
//cuPrintf("\nthread %d(idy) going to loop %d(i)\n", idy, i);
if(idy >= i){
float t[MAXSIZE+10];
//perform calculation
for(j=0; j<size+1; j++){
if(j >= i-1){
t[j] = temp[i-1][j] - (temp[i-1][i-1] / temp[idy][i-1]) * temp[idy][j];
//cuPrintf("calculate No %d, answer %f\n", j, t);

}
}
__syncthreads();
//store data
for(j=0; j<size+1; j++){
if(j >= i-1){
temp[idy][j] = t[j];
}
}
}
__syncthreads();
}

//copy to host
for(i=0; i<size+1; i++){
b_d[idy * (size+1) + i] = temp[idy][i];
}
}