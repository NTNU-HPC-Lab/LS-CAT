#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int locationAlgo(double *x, double xadv, unsigned int nx)
{
unsigned int location = 0;
while (x[location] < xadv && location < nx)
location++;
if(location == 0)
return location;
else
return location-1;
}
__global__ void find_advection_point_location_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int nx, unsigned int ny, unsigned int *cellx, unsigned int *celly, unsigned int *tracker, double xlim1, double xlim2, double ylim1, double ylim2, unsigned int TileSize)
{
unsigned int bx = blockIdx.x;
unsigned int by = blockIdx.y;

unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;

unsigned int index_x = bx * TileSize + tx;
unsigned int index_y = by * TileSize + ty;

unsigned int indexToWrite = index_y * nx + index_x;

bool xoutofbounds = false;
bool youtofbounds = false;

if(!((xadv[indexToWrite] > xlim1) && (xadv[indexToWrite] < xlim2)))
xoutofbounds = true;
if(!((yadv[indexToWrite] > ylim1) && (yadv[indexToWrite] < ylim2)))
youtofbounds = true;

if(!xoutofbounds && !youtofbounds)
{
tracker[indexToWrite] = 1;
cellx[indexToWrite] = locationAlgo(x,xadv[indexToWrite],nx);
celly[indexToWrite] = locationAlgo(y,yadv[indexToWrite],ny);
}
else
if(!xoutofbounds && youtofbounds)
{
tracker[indexToWrite] = 2;
cellx[indexToWrite] = locationAlgo(x,xadv[indexToWrite],nx);
if(yadv[indexToWrite] <= ylim1)
celly[indexToWrite] = 0;
else
if(yadv[indexToWrite] >= ylim2)
celly[indexToWrite] = ny-2;
}
else
if(xoutofbounds && !youtofbounds)
{
tracker[indexToWrite] = 3;
celly[indexToWrite] = locationAlgo(y,yadv[indexToWrite],ny);
if(xadv[indexToWrite] <= xlim1)
cellx[indexToWrite] = 0;
else
if(xadv[indexToWrite] >= xlim2)
cellx[indexToWrite] = nx-2;
}
else
if(xoutofbounds && youtofbounds)
tracker[indexToWrite] = 4;
}