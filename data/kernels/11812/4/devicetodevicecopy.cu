#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devicetodevicecopy(double *dphi, double *dpsix, double *dpsiy, double *mphi, double *mpsix, double *mpsiy, unsigned int nx, unsigned int TileSize)
{
unsigned int bx = blockIdx.x;
unsigned int by = blockIdx.y;

unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;

unsigned int index_x = bx * TileSize + tx;
unsigned int index_y = by * TileSize + ty;

unsigned int indexToWrite = index_y * nx + index_x;

mphi[indexToWrite] = dphi[indexToWrite];
mpsix[indexToWrite] = dpsix[indexToWrite];
mpsiy[indexToWrite] = dpsiy[indexToWrite];
}