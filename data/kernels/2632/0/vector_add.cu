#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_add(float *out, float *a, float *b, int n) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for(int i = index; i < n; i += stride){
out[i] = a[i] + b[i];
}
}