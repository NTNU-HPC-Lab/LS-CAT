#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_add(float *out, float *a, float *b, int n) {
int index = threadIdx.x;
int stride = blockDim.x;
for(int i = index; i < n; i += stride){
out[i] = a[i] + b[i];
}
}