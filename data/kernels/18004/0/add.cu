#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
int index=threadIdx.x+blockIdx.x*blockDim.x;
if(index<SIZE)
{
c[index] = a[index] + b[index];
}
}