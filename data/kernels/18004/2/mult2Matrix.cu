#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult2Matrix(float *M, float *N, float *P) {
__shared__ int shared_m_tile[TILE_WIDTH][TILE_WIDTH];
__shared__ int shared_n_tile[TILE_WIDTH][TILE_WIDTH];

int tx = threadIdx.x;
int ty = threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
//check if thread directly maps to the dimensions of the resulting matrix
if (row < WIDTH && col < WIDTH)
{
float result = 0;
int k;
int phase;
//calculate P matrix indexes in phases. Each phase shares
//TILE_SIZE * TILE_SIZE data copied to the shared matrix M
//and matrix N.
for (phase = 0; phase <= WIDTH / TILE_WIDTH; phase++)
{
shared_m_tile[ty][tx] = M[row * WIDTH + phase * TILE_WIDTH + tx];
shared_n_tile[ty][tx] = N[(phase * TILE_WIDTH + ty) * WIDTH + col];
__syncthreads();

for (k = 0; k < TILE_WIDTH; k++)
{
if (k + (phase * TILE_WIDTH) < WIDTH)
{
result += (shared_m_tile[ty][k] * shared_n_tile[k][tx]);
}
}
__syncthreads();
}
P[row * WIDTH + col] = result;
}
}