#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Blurrig(unsigned char* output, unsigned char* input, int height, int width) {
int Col = threadIdx.x + blockIdx.x * blockDim.x;
int Row = threadIdx.y + blockIdx.y * blockDim.y;

if (Col < width && Row < height) {
int pixVal = 0;
int pixels = 0;
for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow)
{
for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol)
{
int curRow = Row + blurRow;
int curCol = Col + blurCol;
//verify we have a valid image pixel
if (curRow > -1 && curRow<height && curCol>-1 && curCol < width) {
pixVal += input[curRow * width + curCol];
pixels++;//keep track of number of pixels in the avg
}
}
}
//write our new pixel value
output[Row * width + Col] = (unsigned char)(pixVal / pixels);
}
}