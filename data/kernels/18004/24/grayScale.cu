#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale(unsigned char* imgInput, unsigned char* imgOutput, int Row, int Col) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

if ((row < Col) && (col < Row)) {
imgOutput[row * Row + col] = imgInput[(row * Row + col) * 3 + 2] * 0.299 + imgInput[(row * Row + col) * 3 + 1] * 0.587 + imgInput[(row * Row + col) * 3] * 0.114;
}
}