#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void greyConvertor(unsigned char* output, uchar3 const* input, const uint height, const uint width) {
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

if (x < width && y < height) {
int grayOffset = y*width + x;
unsigned char r = input[grayOffset].x;
unsigned char g = input[grayOffset].y;
unsigned char b = input[grayOffset].z;
output[grayOffset] = 0.21f*r + 0.72f*g + 0.07f*b;
}
}