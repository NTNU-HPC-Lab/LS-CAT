#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolve(unsigned char* imgInput, float* mask, int width, int height, int paddingX, int paddingY, int kWidth, int kHeight, unsigned char* imgOutput)
{
// Calculate our pixel's location
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

float sum = 0.0;
int   pWidth = kWidth / 2;
int   pHeight = kHeight / 2;

//Solo ejecuta validos pixeles
if (x >= pWidth + paddingX && y >= pHeight + paddingY && x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
y < (blockDim.y * gridDim.y) - pHeight - paddingY)
{
for (int j = -pHeight; j <= pHeight; j++)
{
for (int i = -pWidth; i <= pWidth; i++)
{
// Sample the weight for this location
int ki = (i + pWidth);
int kj = (j + pHeight);
float w = mask[(kj * kWidth) + ki];


sum += w * float(imgInput[((y + j) * width) + (x + i)]);
}
}
}
imgOutput[(y * width) + x] = (unsigned char)sum;
}