#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult2Matrix(float *M, float *N, float *P) {
// Calculate the row index of the P element and M
int Row = blockIdx.y * blockDim.y + threadIdx.y;
// Calculate the column index of P and N
int Col = blockIdx.x * blockDim.x + threadIdx.x;
if ((Row < WIDTH) && (Col < WIDTH)) {
float Pvalue = 0;
// each thread computes one element of the block sub-matrix
for (int k = 0; k < WIDTH; ++k) {
Pvalue += M[Row*WIDTH + k] * N[k*WIDTH + Col];
}
P[Row*WIDTH + Col] = Pvalue;
}
}