#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Convolution_2D_globalMemory(unsigned char* imgInput,unsigned char* imgOutput, const float* mask, int height, int width, int channels) {

int Row, Col, filterRow, filterCol;

int rows = threadIdx.x + blockIdx.x * blockDim.x;
int cols = threadIdx.y + blockIdx.y * blockDim.y;
float sum = 0;

Row = rows - MASK_WIDTH /2;
Col = cols - MASK_WIDTH /2;
for (int c = 0; c < channels; c++)
{
sum = 0;
for (int i = 0; i < MASK_WIDTH; i++)
{
for (int j = 0; j < MASK_WIDTH; j++)
{
filterRow = Row + i;
filterCol = Col + j;

if ((filterRow >= 0) && (filterRow < height) && (filterCol >= 0) && (filterCol < width))
{
sum += imgInput[(filterRow * height + filterCol) * channels + c] * mask[i * MASK_WIDTH + j];
}
else { sum = 0; }
}
}
sum/=MASK_WIDTH *MASK_WIDTH ;
imgOutput[(rows * width + cols) * channels + c] = (unsigned char)sum;
}
}