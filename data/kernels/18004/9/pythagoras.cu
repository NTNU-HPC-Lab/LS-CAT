#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pythagoras(unsigned char* Gx, unsigned char* Gy, unsigned char* G, unsigned char* theta)
{
int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

float af = float(Gx[idx]);
float bf = float(Gy[idx]);

G[idx] = (unsigned char)sqrtf(af * af + bf * bf);
theta[idx] = (unsigned char)atan2f(af, bf)*63.994;

}