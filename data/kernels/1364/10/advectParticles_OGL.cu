#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
int p;

// gtidx is the domain location in x for this thread
float2 pterm, vterm;
if (gtidx < dx) {
for (p = 0; p < lb; p++) {
// fi is the domain location in y for this thread
int fi = gtidy + p;
if (fi < dy) {
int fj = fi * dx + gtidx;
pterm = part[fj];

int xvi = ((int)(pterm.x * dx));
int yvi = ((int)(pterm.y * dy));
vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

pterm.x += dt * vterm.x;
pterm.x = pterm.x - (int)pterm.x;
pterm.x += 1.f;
pterm.x = pterm.x - (int)pterm.x;
pterm.y += dt * vterm.y;
pterm.y = pterm.y - (int)pterm.y;
pterm.y += 1.f;
pterm.y = pterm.y - (int)pterm.y;

part[fj] = pterm;
}
} // If this thread is inside the domain in Y
} // If this thread is inside the domain in X
}