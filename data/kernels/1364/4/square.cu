#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(int *array, int arrayCount)
{
extern __shared__ int dynamicSmem[];
int idx = threadIdx.x + blockIdx.x * blockDim.x;

if (idx < arrayCount) {
array[idx] *= array[idx];
}
}