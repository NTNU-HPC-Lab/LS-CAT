#include "hip/hip_runtime.h"
#include "includes.h"
//============================================================================
// Name        : CudaMap.cu
// Author      : Hang
//============================================================================



using namespace std;


__global__ void addTen(float* d, int count) {

int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;

// Thread position in the block
int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y +
blockDim.x * blockDim.y * threadIdx.z;

// Block position in grid
int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y +
gridDim.x * gridDim.y * blockIdx.z;

// Final thread ID
int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

if (tid < count) {
d[tid] = d[tid] + 10;
}

}