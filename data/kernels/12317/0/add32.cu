#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add32(float* A, float *B, int size) {
int block = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
int index = block * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
if(index >= size) return;

A[index] = A[index] + B[index];
}