#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000
#define MAX_ERR 1e-6


__global__ void vector_add(float* out,float* a,float* b,int n){
int index = threadIdx.x;
int stride = blockDim.x;
for(int i=index ; i<n ;i=i+stride){
out[i]=a[i]+b[i];
}
}