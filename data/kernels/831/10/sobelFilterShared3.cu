#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelFilterShared3(unsigned char* g_DataIn, unsigned char * g_DataOut, unsigned int width, unsigned int height){
__shared__ char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

int x = blockIdx.x * TILE_WIDTH + threadIdx.x - FILTER_RADIUS;
int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - FILTER_RADIUS;

//Clamp to the center
x = max(FILTER_RADIUS, x);
x = min(x, width - FILTER_RADIUS - 1);
y = max(FILTER_RADIUS, y);
y = min(y, height - FILTER_RADIUS - 1);

unsigned int index = y * width + x;
unsigned int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;

sharedMem[sharedIndex] = g_DataIn[index];

__syncthreads();

if(		threadIdx.x >= FILTER_RADIUS && threadIdx.x < BLOCK_WIDTH - FILTER_RADIUS
&&	threadIdx.y >= FILTER_RADIUS && threadIdx.y < BLOCK_HEIGHT - FILTER_RADIUS)
{
int sum = 0;

for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; ++dy)
for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; ++dx)
{
int pixelValue = (int)(sharedMem[sharedIndex + (dy * blockDim.x + dx)]);

sum += pixelValue;
}

g_DataOut[index] = (unsigned char)(sum / FILTER_AREA);
}
}