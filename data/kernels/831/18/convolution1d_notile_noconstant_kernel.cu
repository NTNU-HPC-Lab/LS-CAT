#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution1d_notile_noconstant_kernel(int *In, int *Out){
unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
int Value = 0;
int N_start_point = index - (Mask_size/2);
for ( int j = 0; j  < Mask_size; j ++) {
if (N_start_point + j >= 0 && N_start_point + j < N_elements) {
Value += In[N_start_point + j] * Global_Mask[j];
}
}
Out[index] = Value;
}