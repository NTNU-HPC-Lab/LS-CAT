#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelFilterShared2(unsigned char *data, unsigned char *result, int width, int height){
// Data cache: threadIdx.x , threadIdx.y
int ty = threadIdx.y;
int tx = threadIdx.x;

// shared memory represented here by 1D array
// each thread loads two values from global memory into shared mem
const int n = Mask_size / 2;
__shared__ int s_data[BLOCKSIZE * (BLOCKSIZE + Mask_size * 2)];

// global mem address of the current thread in the whole grid
const int pos = tx + blockIdx.x * blockDim.x + ty * width + blockIdx.y * blockDim.y * width;

// load cache (32x32 shared memory, 16x16 threads blocks)
// each threads loads four values from global memory into shared mem
// if in image area, get value in global mem, else 0
int y; // image based coordinate

// original image based coordinate
const int y0 = ty + blockIdx.y * blockDim.y;
const int shift = ty * (BLOCKSIZE);

// case1: upper left
y = y0 - n;
if ( y < 0 )
s_data[tx + shift] = 0;
else
s_data[tx + shift] = data[ pos - (width * n)];

// case2: lower
y = y0 - n;
const int shift1 = shift + blockDim.y * BLOCKSIZE;

if ( y > height - 1)
s_data[tx + shift1] = 0;
else
s_data[tx + shift1] = data[pos +  (width * n)];

__syncthreads();

// convolution
int sum = 0;
for (int i = 0; i <= n*2; i++)
sum += s_data[tx + (ty+i) * BLOCKSIZE] * Global_Mask[i];

result[pos] = sum;
}