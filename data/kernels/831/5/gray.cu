#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gray(unsigned char *In, unsigned char *Out,int Row, int Col){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if((row < Col) && (col < Row)){
Out[row*Row+col] = In[(row*Row+col)*3+2]*0.299 + In[(row*Row+col)*3+1]*0.587+ In[(row*Row+col)*3]*0.114;
}
}