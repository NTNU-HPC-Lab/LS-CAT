#include "hip/hip_runtime.h"
#include "includes.h"
# include <bits/stdc++.h>
# include <hip/hip_runtime.h>

#define SIZE 60// Global Size
#define BLOCK_SIZE 1024
using namespace std;

//::::::::::::::::::::::::::::::::::::::::::GPU::::::::::::::::::::::::::::::::

// :::: Kernel




// :::: Calls
__global__ void kernel_prefix_sum_inefficient(double *g_idata,double *g_odata,int l){ // Sequential Addressing technique

__shared__ double sdata[BLOCK_SIZE];
// each thread loads one element from global to shared mem

unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

if(i<l && tid !=0){
sdata[tid] = g_idata[i-1];
}else{
sdata[tid] = 0;
}

// do reduction in shared mem
for(unsigned int s=1;s<=tid;s *=2){
__syncthreads();
sdata[tid]+=sdata[tid-s];
}

// write result for this block to global mem
g_odata[i] = sdata[tid];
}