#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution1d_constant_simple_kernel(int *In, int *Out){

int i = blockIdx.x*blockDim.x + threadIdx.x;
__shared__ float N_ds[TILE_SIZE];
N_ds[threadIdx.x] = In[i];
__syncthreads();
int This_tile_start_point = blockIdx.x * blockDim.x;
int Next_tile_start_point = (blockIdx.x + 1) * blockDim.x;
int N_start_point = i - (Mask_size/2);
int Pvalue = 0;
for (int j = 0; j < Mask_size; j ++){
int N_index = N_start_point + j;
if (N_index >= 0 && N_index < N_elements){
if ((N_index >= This_tile_start_point) && (N_index < Next_tile_start_point)){
Pvalue += N_ds[threadIdx.x+j-(Mask_size/2)]*Global_Mask[j];
} else{ Pvalue += In[N_index] * Global_Mask[j]; }
}
}
Out[i] = Pvalue;
}