#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelNormalMul(float *Mat1,float *Mat2,float *Mat3,int m,int n,int p){
int j = threadIdx.y + blockDim.y * blockIdx.y; // row
int i = threadIdx.x + blockDim.x * blockIdx.x; // col

if((j<m) && (i<p)){
float value=0.0;
for(int k=0;k<n;++k){
value+=Mat1[n*j+k]*Mat2[p*k+i];
}
Mat3[p*j+i]=value;
}
}