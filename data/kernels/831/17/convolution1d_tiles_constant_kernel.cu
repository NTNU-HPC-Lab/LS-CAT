#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution1d_tiles_constant_kernel(int *In, int *Out){
unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
__shared__ int Tile[TILE_SIZE + Mask_size - 1];
int n = Mask_size/2;
int halo_left_index  = (blockIdx.x - 1 ) * blockDim.x + threadIdx.x;
if (threadIdx.x  >= blockDim.x - n ){
Tile[threadIdx.x - (blockDim.x - n )] = (halo_left_index < 0) ? 0 : In[halo_left_index];
}

if(index<N_elements){Tile[n + threadIdx.x] = In[index];
}else{Tile[n + threadIdx.x] = 0;}
int halo_right_index = (blockIdx.x + 1 ) * blockDim.x + threadIdx.x;
if (threadIdx.x < n) {
Tile[n + blockDim.x + threadIdx.x]=  (halo_right_index >= N_elements) ? 0 : In[halo_right_index];
}

__syncthreads();
int Value = 0;
for (unsigned int j = 0; j  < Mask_size; j ++) {
Value += Tile[threadIdx.x + j] * Global_Mask[j];
}
Out[index] = Value;
}