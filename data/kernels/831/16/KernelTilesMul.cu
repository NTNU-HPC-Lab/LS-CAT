#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  KernelTilesMul(float *Mat1,float *Mat2,float *Mat3,int rowM1,int colM1,int colM2){

__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

float Pvalue = 0.0;


for(int k = 0; k < (colM1+TILE_WIDTH-1)/(TILE_WIDTH); ++k){

if(k*TILE_WIDTH + tx < colM1 && row < rowM1){
Mds[ty][tx] = Mat1[row*colM1 + k*TILE_WIDTH + tx];
}else{
Mds[ty][tx] = 0.0;
}
if(k*TILE_WIDTH + ty < colM1 && col < colM2){
Nds[ty][tx] = Mat2[(k*TILE_WIDTH + ty) * colM2 + col];
}else{
Nds[ty][tx] =0.0;
}

__syncthreads();

for(int k = 0; k < TILE_WIDTH; ++k){
Pvalue += Mds[ty][k] * Nds[k][tx];
}
__syncthreads();
}

if (row < rowM1 && col < colM2){
Mat3[row*colM2+col] = Pvalue;
}

}