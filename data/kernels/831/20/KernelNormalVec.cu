#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelNormalVec(double *g_idata,double *g_odata,int l){ // Sequential Addressing technique

__shared__ double sdata[BLOCK_SIZE];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<l){ // bad thing -> severely punished performance.
sdata[tid] = g_idata[i];
}else{
sdata[tid] = 0.0;
}

__syncthreads();
// do reduction in shared mem
for(unsigned int s=blockDim.x/2; s > 0; s >>= 1) {
if(tid < s){
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}
// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}