#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelFilterShared(unsigned char *data, unsigned char *result, int width, int height){
// Data cache: threadIdx.x , threadIdx.y
const int n = Mask_size / 2;
__shared__ int s_data[BLOCKSIZE + Mask_size * 2 ][BLOCKSIZE + Mask_size * 2];

// global mem address of the current thread in the whole grid
const int pos = threadIdx.x + blockIdx.x * blockDim.x + threadIdx.y * width + blockIdx.y * blockDim.y * width;

// load cache (32x32 shared memory, 16x16 threads blocks)
// each threads loads four values from global memory into shared mem
// if in image area, get value in global mem, else 0
int x, y; // image based coordinate

// original image based coordinate
const int x0 = threadIdx.x + blockIdx.x * blockDim.x;
const int y0 = threadIdx.y + blockIdx.y * blockDim.y;

// case1: upper left
x = x0 - n;
y = y0 - n;
if ( x < 0 || y < 0 )
s_data[threadIdx.y][threadIdx.x] = 0;
else
s_data[threadIdx.y][threadIdx.x] = *(data + pos - n - (width * n));

// case2: upper right
x = x0 + n;
y = y0 - n;
if ( x > (width - 1) || y < 0 )
s_data[threadIdx.y][threadIdx.x + blockDim.x] = 0;
else
s_data[threadIdx.y][threadIdx.x + blockDim.x] = *(data + pos + n - (width * n));

// case3: lower left
x = x0 - n;
y = y0 + n;
if (x < 0 || y > (height - 1))
s_data[threadIdx.y + blockDim.y][threadIdx.x] = 0;
else
s_data[threadIdx.y + blockDim.y][threadIdx.x] = *(data + pos - n + (width * n));

// case4: lower right
x = x0 + n;
y = y0 + n;
if ( x > (width - 1) || y > (height - 1))
s_data[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = 0;
else
s_data[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = *(data + pos + n + (width * n));

__syncthreads();

// convolution
int sum = 0;
x = n + threadIdx.x;
y = n + threadIdx.y;
for (int i = - n; i <= n; i++)
for (int j = - n; j <= n; j++)
sum += s_data[y + i][x + j] * Global_Mask[n + i] * Global_Mask[n + j];

result[pos] = sum;
}