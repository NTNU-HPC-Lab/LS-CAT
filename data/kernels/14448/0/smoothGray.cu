#include "hip/hip_runtime.h"
#include "includes.h"










__global__ void smoothGray (unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas)
{
unsigned int indice = (blockIdx.y * blockDim.x * 65536) + (blockIdx.x * 1024) + threadIdx.x; // calcula o indice do vetor com base nas dimensões de bloco e indice da thread
if(indice >= cols*linhas)
return;
//indices para o campo da imagem que participará do smooth
int i_begin = (indice/(int)cols) - 2, i_end = (indice/(int)cols)+2;
int j_begin = (indice%(int)cols) - 2, j_end = (indice%(int)cols)+2;
if(i_begin<0) i_begin = 0;
if(j_begin<0) j_begin = 0;
if(i_end>=cols) i_end = cols-1;
if(j_end>=cols) j_end = cols-1;

//calcula o smooth no ponto de indice da thread
int media = 0;
int qtd = 0;
for (int i = i_begin; i<=  i_end; ++i)
{
for(int j = j_begin; j<= j_end; ++j)
{
media += imagem[(i*cols)+j];
qtd++;
}
}
saida[indice] = (unsigned char)(media/qtd);
}