#include "hip/hip_runtime.h"
#include "includes.h"










__global__ void smoothColor (unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas)
{
unsigned int indice = (blockIdx.y * blockDim.x * 65536) + (blockIdx.x * 1024) + threadIdx.x; // calcula o indice do vetor com base nas dimensões de bloco e indice da thread
if(indice >= cols*linhas)
return;
//indices para o campo da imagem que participará do smooth
int i_begin = (indice/(int)cols)-2, i_end = (indice/(int)cols)+2;
int j_begin = (indice%(int)cols)-2, j_end = (indice%(int)cols)+2;
if(i_begin<0) i_begin = 0;
if(j_begin<0) j_begin = 0;
if(i_end>=cols) i_end = cols-1;
if(j_end>=cols) j_end = cols-1;

//calcula o smooth no ponto de indice da thread
int media[3] = {0,0,0};
int qtd = 0;
for (int i = i_begin; i<=  i_end; ++i)
{
for(int j = j_begin; j<= j_end; ++j)
{
media[0] += imagem[((i*cols)+j)*3];
media[1] += imagem[((i*cols)+j)*3+1];
media[2] += imagem[((i*cols)+j)*3+2];
qtd++;
}
}

saida[indice*3] = (unsigned char)(media[0]/qtd);
saida[indice*3+1] = (unsigned char)(media[1]/qtd);
saida[indice*3+2] = (unsigned char)(media[2]/qtd);
}