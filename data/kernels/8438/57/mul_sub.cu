#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < in1ScalarCount; tid += stride) {
out[tid] = in1[tid] * in2[tid % in2ScalarCount];
}
}