#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride) {
float temp = d[tid];
if (temp > gradClip) temp = gradClip;
if (temp < -gradClip) temp = -gradClip;
m[tid] *= momentum;
m[tid] += temp;
if (nesterov) { temp += momentum * m[tid]; }
else { temp = m[tid]; }
x[tid] -= learning_rate * temp;
d[tid] = 0;
}
}