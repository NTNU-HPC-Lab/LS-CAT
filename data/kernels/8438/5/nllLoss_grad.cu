#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int offset = tid * x_stride + target[tid];
xGrad[offset] += -1 * yGrad[tid];
}