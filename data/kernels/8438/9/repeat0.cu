#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < outScalarCount; tid += stride) {
int linearIndex = tid;
int outIndex0 = linearIndex / outStride0;
linearIndex = linearIndex - outIndex0 * outStride0;
int outIndex1 = linearIndex / outStride1;
int outIndex2 = linearIndex - outIndex1 * outStride1;
int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
out[tid] = in[inIndex];
}
}