#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void arrayFill_greg(float* data, float value, int size) {
int stride = gridDim.x * blockDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
for (int i = tid; i < size; i += stride) data[i] = value;
}