#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int offset = tid * x_stride + target[tid];
y[tid] = -1 * x[offset];
}