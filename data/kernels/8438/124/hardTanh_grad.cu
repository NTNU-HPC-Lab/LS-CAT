#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = gridDim.x * blockDim.x;
for (int i = tid; i < size; i += stride) {
if (inplace) {
if (in_x[i] < min_val || in_x[i] > max_val) in_d[i] = 0;
} else {
if (in_x[i] >= min_val && in_x[i] <= max_val) in_d[i] += out_d[i];
}
}
}