#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < in1ScalarCount; tid += stride) {
out[tid] = in1[tid] * in2[tid % in2ScalarCount];
}
}