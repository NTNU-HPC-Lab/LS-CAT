#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < in1ScalarCount; tid += stride) {
int index = tid % in2ScalarCount;
in1_d[tid] += out[tid] * in2_x[index];
in2_d[tid] = in1_x[tid] * out[tid];  // this is the temp array, need to be reduced!
}
}