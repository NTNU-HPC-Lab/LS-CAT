#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in1[tid] - in2[tid];
}