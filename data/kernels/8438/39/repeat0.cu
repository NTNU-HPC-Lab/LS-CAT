#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < outScalarCount; tid += stride) {
int linearIndex = tid;
int outIndex0 = linearIndex / outStride0;
linearIndex = linearIndex - outIndex0 * outStride0;
int outIndex1 = linearIndex / outStride1;
int outIndex2 = linearIndex - outIndex1 * outStride1;
int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
out[tid] = in[inIndex];
}
}