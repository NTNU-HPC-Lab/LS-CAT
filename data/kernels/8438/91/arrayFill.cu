#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayFill(float* data, float value, int size) {
int stride = gridDim.x * blockDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
for (int i = tid; i < size; i += stride) data[i] = value;
}