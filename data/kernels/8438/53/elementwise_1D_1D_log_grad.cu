#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) in_d[tid] += out_d[tid] / in_x[tid];
}