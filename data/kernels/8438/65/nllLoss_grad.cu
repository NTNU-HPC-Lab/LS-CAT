#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int offset = tid * x_stride + target[tid];
xGrad[offset] += -1 * yGrad[tid];
}