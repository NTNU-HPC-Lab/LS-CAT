#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void sum_grad(float* in, int inSize0, int inSize1, int inSize2, int inSize3, int nElement, float* out, int outStride0, int outStride1, int outStride2, int dim) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (int i = tid; i < nElement; i += stride) {
int inOff2 = i / inSize3;
int inDim3 = i - inOff2 * inSize3;
int inOff1 = inOff2 / inSize2;
int inDim2 = inOff2 - inOff1 * inSize2;
int inDim0 = inOff1 / inSize1;
int inDim1 = inOff1 - inDim0 * inSize1;
int outOff = 0;
if (dim == 0) outOff = inDim1 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
if (dim == 1) outOff = inDim0 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
if (dim == 2) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim3 * outStride2;
if (dim == 3) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim2 * outStride2;
in[i] += out[outOff];
}
}