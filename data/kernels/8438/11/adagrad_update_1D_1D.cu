#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride) {
if (d[tid] > clip) d[tid] = clip;
if (d[tid] < -clip) d[tid] = -clip;
m[tid] += d[tid] * d[tid];
x[tid] -= lr * d[tid] / sqrt(m[tid] + 0.00000001);
d[tid] = 0;
}
}