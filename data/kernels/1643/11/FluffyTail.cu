#include "hip/hip_runtime.h"
#include "includes.h"
// Cuckarood Cycle, a memory-hard proof-of-work by John Tromp and team Grin
// Copyright (c) 2018 Jiri Photon Vadura and John Tromp
// This GGM miner file is covered by the FAIR MINING license

//Includes for IntelliSense
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif





typedef uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef uint64_t u64;

typedef u32 node_t;
typedef u64 nonce_t;


#define DUCK_SIZE_A 134LL
#define DUCK_SIZE_B 86LL

#define DUCK_A_EDGES (DUCK_SIZE_A * 1024LL)
#define DUCK_A_EDGES_64 (DUCK_A_EDGES * 64LL)

#define DUCK_B_EDGES (DUCK_SIZE_B * 1024LL)
#define DUCK_B_EDGES_64 (DUCK_B_EDGES * 64LL)

#define EDGE_BLOCK_SIZE (64)
#define EDGE_BLOCK_MASK (EDGE_BLOCK_SIZE - 1)

#define EDGEBITS 29
#define NEDGES2 ((node_t)1 << EDGEBITS)
#define NEDGES1 (NEDGES2/2)
#define NNODES1 NEDGES1
#define NNODES2 NEDGES2

#define EDGEMASK (NEDGES2 - 1)
#define NODE1MASK (NNODES1 - 1)

#define CTHREADS 1024
#define CTHREADS512 512
#define BKTMASK4K (4096-1)
#define BKTGRAN 64

#define EDGECNT 562036736
#define BUKETS 4096
#define BUKET_MASK (BUKETS-1)
#define BUKET_SIZE (EDGECNT/BUKETS)

#define XBITS 6
const u32 NX = 1 << XBITS;
const u32 NX2 = NX * NX;
const u32 XMASK = NX - 1;
const u32 YBITS = XBITS;
const u32 NY = 1 << YBITS;
const u32 YZBITS = EDGEBITS - XBITS;
const u32 ZBITS = YZBITS - YBITS;
const u32 NZ = 1 << ZBITS;
const u32 ZMASK = NZ - 1;

#define ROTL(x,b) ( ((x) << (b)) | ( (x) >> (64 - (b))) )
#define SIPROUND \
{ \
v0 += v1; v2 += v3; v1 = ROTL(v1,13); \
v3 = ROTL(v3,16); v1 ^= v0; v3 ^= v2; \
v0 = ROTL(v0,32); v2 += v1; v0 += v3; \
v1 = ROTL(v1,17);   v3 = ROTL(v3,25); \
v1 ^= v2; v3 ^= v0; v2 = ROTL(v2,32); \
}
__global__  void FluffyTail(const uint2 * source, uint2 * destination, const int * sourceIndexes, int * destinationIndexes)
{
const int lid = threadIdx.x;
const int group = blockIdx.x;

int myEdges = sourceIndexes[group];
__shared__ int destIdx;

if (lid == 0)
destIdx = atomicAdd(destinationIndexes, myEdges);

__syncthreads();

if (lid < myEdges)
{
destination[destIdx + lid] = source[group * DUCK_B_EDGES / 4 + lid];
}
}