#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
int idx =  blockIdx.x * blockDim.x + threadIdx.x;
C[idx] = A[idx] + B[idx];
}