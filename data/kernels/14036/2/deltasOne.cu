#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void deltasOne(float *inputs, float *outputs, float *weights, float *weightsDeltas, int offsetInputImages, int inputSize){
int gid = blockIdx.x * blockDim.x + threadIdx.x;
float sum=0;
int offsetDeltas = (inputSize+1)*gid;
int offsetInput = inputSize*(gid+offsetInputImages);

for(int imageIndex=0;imageIndex<=inputSize;imageIndex++){
weightsDeltas[offsetDeltas+imageIndex]=0;
}

for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
sum+=inputs[offsetInput+imageIndex]*weights[imageIndex];
}
sum+=weights[inputSize];
if(sum>0) sum=1;
else sum=0;
sum=outputs[offsetInputImages+gid]-sum;
if(sum!=0){
for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
weightsDeltas[offsetDeltas+imageIndex]+=sum*inputs[offsetInput+imageIndex];
}
weightsDeltas[offsetDeltas+inputSize]+=sum;
}

}