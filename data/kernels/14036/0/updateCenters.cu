#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void updateCenters(float *centers, float *images, int *updates, int noClusters)
{
int gid = blockIdx.x * blockDim.x + threadIdx.x;
int imagesOffset;
int centersIndex=0;

float sum=0;
int index=0;
float weight;
float min;
int minCenterIndex=-1;
int imageSize=784;
float pImage[784];

imagesOffset = gid*imageSize;
for (index=0;index<imageSize;index++){
pImage[index]=images[imagesOffset+index];
}


min=100000000;
for(centersIndex=0;centersIndex<100;centersIndex++)
{
sum = 0;
for(index=0;index<784;index++)
{
weight = centers[centersIndex*imageSize+index]-pImage[index];
sum = sum+weight*weight;
}
if (sum<min)
{
min = sum;
minCenterIndex = centersIndex;
}
}
updates[gid]=minCenterIndex;
}