#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void deltasBatch(float *inputs, float *outputs, float *weights, float *weightsDeltas, int noInputs, int inputSize){
int gid = blockIdx.x * blockDim.x + threadIdx.x;
float sum=0;
int offsetDeltas = (inputSize+1)*gid;
int offsetInput = noInputs*inputSize*gid;
int offsetOutputs = noInputs*gid;

for(int imageIndex=0;imageIndex<=inputSize;imageIndex++){
weightsDeltas[offsetDeltas+imageIndex]=0;
}

for (int i=0;i<noInputs;i++){
sum=0;
for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
sum+=inputs[offsetInput+i*inputSize+imageIndex]*weights[imageIndex];
}
sum+=weights[inputSize];
if(sum>0) sum=1;
else sum=0;
sum=outputs[offsetOutputs+i]-sum;
if(sum!=0){
for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
weightsDeltas[offsetDeltas+imageIndex]+=sum*inputs[offsetInput+i*inputSize+imageIndex];
}
weightsDeltas[offsetDeltas+inputSize]+=sum;
}
}
}