#include "hip/hip_runtime.h"
#include "includes.h"
#define NO_HIDDEN_NEURONS 5
extern "C"
__global__ void deltasBatch(float *inputs, float *outputs, float *weights, float *weightsDeltas, int noInputs, int inputSize){
int gid = blockIdx.x * blockDim.x + threadIdx.x;
float sum=0;
int offsetDeltas = ((inputSize+1)*NO_HIDDEN_NEURONS+NO_HIDDEN_NEURONS+1)*gid;
int offsetInput = noInputs*inputSize*gid;
int offsetOutputs = noInputs*gid;
float activationHidden[NO_HIDDEN_NEURONS];
float error;

for(int hidden=0;hidden<NO_HIDDEN_NEURONS;hidden++){
for(int imageIndex=0;imageIndex<=inputSize;imageIndex++){
weightsDeltas[offsetDeltas+(inputSize+1)*hidden+imageIndex]=0;
}
}

for(int hidden=0;hidden<=NO_HIDDEN_NEURONS;hidden++){
weightsDeltas[offsetDeltas+(inputSize+1)*NO_HIDDEN_NEURONS+hidden]=0;
}

for (int i=0;i<noInputs;i++){
for(int hidden=0;hidden<NO_HIDDEN_NEURONS;hidden++){
sum=0;
for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
sum+=inputs[offsetInput+i*inputSize+imageIndex]*weights[(inputSize+1)*hidden+imageIndex];
}
sum+=weights[(inputSize+1)*hidden+inputSize];
if(sum>0) activationHidden[hidden]=1;
else  activationHidden[hidden]=0;
//activationHidden[hidden]=sum/(1+abs(sum));
}
sum=0;
for(int hidden=0;hidden<NO_HIDDEN_NEURONS;hidden++){
sum+=activationHidden[hidden]*weights[(inputSize+1)*NO_HIDDEN_NEURONS+hidden];
}
sum+=weights[(inputSize+1)*NO_HIDDEN_NEURONS+NO_HIDDEN_NEURONS];
if(sum>0)sum=1;
else  sum=0;
sum=outputs[offsetOutputs+i]-sum;
if(sum!=0){
for(int hidden=0;hidden<NO_HIDDEN_NEURONS;hidden++){
weightsDeltas[offsetDeltas+(inputSize+1)*NO_HIDDEN_NEURONS+hidden]+=sum*activationHidden[hidden];
}
weightsDeltas[offsetDeltas+(inputSize+1)*NO_HIDDEN_NEURONS+NO_HIDDEN_NEURONS]+=sum;

for(int hidden=0;hidden<NO_HIDDEN_NEURONS;hidden++){
error=sum*weights[(inputSize+1)*NO_HIDDEN_NEURONS+hidden];

if(error>0)error=1;
else error=0;

error=error-activationHidden[hidden];
if(error!=0){
for(int imageIndex=0;imageIndex<inputSize;imageIndex++){
weightsDeltas[offsetDeltas+(inputSize+1)*hidden+imageIndex]+=error*inputs[offsetInput+i*inputSize+imageIndex];
}
weightsDeltas[offsetDeltas+(inputSize+1)*hidden+inputSize]+=error;
}
}
}
}
}