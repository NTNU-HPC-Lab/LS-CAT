#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define TILE_WIDTH 2



// main fn
__global__ void MatrixMult(int m, int n, int k, float *a, float *b, float *c)
{

int row = threadIdx.y + blockIdx.y*blockDim.y;
int col = threadIdx.x + blockIdx.x*blockDim.x;

if((row < m) && (col < k))
{
float temp = 0.0;
for (int i = 0; i < n; ++i)
{
temp += a[row*n+i]*b[col+i*k];
}
c[row*k+col] = temp;
}

}