#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int size = blockDim.x * gridDim.x;

for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
}
}