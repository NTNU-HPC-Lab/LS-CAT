#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_grad(float *output, float *grad, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int size = blockDim.x * gridDim.x;

for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
output[idx] += dt * grad[idx];
}
}