#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activation_function(float x)
{
return 1 / (1 + exp(-x));
}
__global__ void apply_activation_function(float *input, float *output, const int N)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;
const int size = blockDim.x * gridDim.x;

for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
output[idx] = activation_function(input[idx]);
}
}