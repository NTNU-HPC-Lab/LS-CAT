#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_RNG_setup ( hiprandState * state, unsigned long seed, int N )
{
int id = blockIdx.x * blockDim.x + threadIdx.x;

while(id < N) {

hiprand_init( (seed << 20) + id, 0, 0, &state[id]);

id += blockDim.x*gridDim.x;
}
}