#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_totalTemp_kernel ( int N, double * partialT, double * totalT)
{
extern __shared__ double T_cache[];
int tid = threadIdx.x;

T_cache[tid] = partialT[tid];

__syncthreads();

int nTotalThreads = blockDim.x;               /// Total number of active threads

/** Algoritme per calcular la reduccio
*  dels valors actuals a la cache del block */
while(nTotalThreads > 1)
{
int halfPoint = (nTotalThreads >> 1);       /// divide by two, only the first half of the threads will be active.

if (threadIdx.x < halfPoint)
T_cache[threadIdx.x] += T_cache[threadIdx.x + halfPoint];

__syncthreads();                /// imprescindible

nTotalThreads = halfPoint;      /// Reducing the binary tree size by two:
}


/// El primer thread de cada block es el k s'encarrega de fer els calculs finals
if(threadIdx.x == 0) {

double T = T_cache[0];

T /= (kb * dim * N);  /// Instantaneous temperature using the Equipartition Theorem. The kinetic energy is just K = 3N/2 kT

(*totalT) = T;
}
}