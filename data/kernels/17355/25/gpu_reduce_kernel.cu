#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce_kernel (int N, float * vector, float * sum)
{
extern __shared__ float partialSum[];
int tid = threadIdx.x + blockIdx.x*blockDim.x;

partialSum[threadIdx.x] = 0.f;

__syncthreads();

while(tid < N)
{
partialSum[threadIdx.x] += vector[tid];

tid += blockDim.x*gridDim.x;
}

__syncthreads();

int nTotalThreads = blockDim.x;	/// Total number of active threads

/// Algoritme per calcular la reduccio dels valors actuals a la cache del block
while(nTotalThreads > 1)
{
int halfPoint = (nTotalThreads >> 1);	/// divide by two
/// only the first half of the threads will be active.

if (threadIdx.x < halfPoint)
{
partialSum[threadIdx.x] += partialSum[threadIdx.x + halfPoint];
}

/// imprescindible per les reduccions
__syncthreads();

/// Reducing the binary tree size by two:
nTotalThreads = halfPoint;
}

/// El primer thread del primer block es el k s'encarrega de fer els calculs finals
if(threadIdx.x == 0)
(*sum) = partialSum[0];
}