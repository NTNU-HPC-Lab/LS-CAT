#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Week 3
* Parallel Programming
* 2011-2012
* University of Birmingham
*
* This is a first step towards implementing "parallel reduce".
* Reducing means using an operation to aggregate the values of
* a data type, such an array or a list.
*
* For example, to calculate the sum we aggregate addition:
*     a1 + a2 + a3 + a4 ...
* To calculate the maximum we aggregate the max operation:
*     max (a1, max(a2, max(a3, ...
* Note that the order in which the device map, which is parallel,
* and the host map, which is sequential, will differ, therefore the
* operation needs to be associative.
* Operations such as +, * or max are associative, but function of
* two arguments, in general, are not!
*/




using namespace std;


const int ITERS = 500;




/*
* Reference CPU implementation, taken from http://www.songho.ca/dsp/convolution/convolution.html
*/
__global__ void convolve(float* data_in, float* data_out, float* kernel, int kernelSize, int BLOCK_SIZE)
{
int tx = threadIdx.x;
int bk = blockIdx.x;
int pos = (bk * BLOCK_SIZE) + tx;
data_out[pos] = 0;

for(int i = 0; i < kernelSize; i++){
if(pos - i >= 0) {
data_out[pos] += kernel[i] * data_in[pos - i];
}
}

}