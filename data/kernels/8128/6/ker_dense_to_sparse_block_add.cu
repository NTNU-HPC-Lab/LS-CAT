#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n*bsize)
atomicAdd(trg + idx[id/bsize]*bsize+id%bsize, src[id]);
}