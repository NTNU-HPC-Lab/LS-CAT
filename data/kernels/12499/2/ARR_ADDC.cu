#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ARR_ADDC(float* result, float* in1, float* in2, int N)
{
int index = blockDim.x * blockIdx.x + threadIdx.x;
if (index < N)
{
result[index] = in1[index] + in2[index];
}
}