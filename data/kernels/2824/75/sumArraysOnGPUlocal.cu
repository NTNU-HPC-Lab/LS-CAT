#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPUlocal(float *A, float *B, float *C, const int N)
{
float local[4];
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i +4 < N) {
for (int j=0; j < 4; j++) local[j] = 2*A[i+j];
C[i] = A[i] + B[i] + local[threadIdx.x%4];

}

}