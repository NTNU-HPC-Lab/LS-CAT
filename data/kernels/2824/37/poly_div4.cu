#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poly_div4(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < N) {
float x = poli[idx];
float y = 5.0/x;
poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+y;
}
}