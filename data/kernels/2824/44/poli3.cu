#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli3(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = poli[idx];

if (idx < N)
poli[idx] = 5 + 5 * x + 5 * x * x + 5 * x * x * x + 5 * x * x * x * x + 5 * x * x * x * x * x;
}