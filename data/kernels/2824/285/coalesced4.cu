#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void coalesced4(float *A, float *C, const int N)
{
int i = (blockIdx.x * blockDim.x + threadIdx.x)*4;

if (i+3 < N) { C[i] = A[i]; C[i+1] = A[i+1];
C[i+2] = A[i+2]; C[i+3] = A[i+3];}
}