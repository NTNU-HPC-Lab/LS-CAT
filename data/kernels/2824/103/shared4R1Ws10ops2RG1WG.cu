#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R1Ws10ops2RG1WG(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)
Smem[threadIdx.x] = A[i];
__syncthreads();

if (i < N) {
C[i] = A[i] + B[i] - A[i]*A[i] + 3*B[i] - 4*A[i]*B[i] + B[i]*B[i]*7- 8+Smem[(threadIdx.x+1)%512]+Smem[(threadIdx.x+2)%512]+Smem[(threadIdx.x+3)%512]+Smem[(threadIdx.x+4)%512];
}
}