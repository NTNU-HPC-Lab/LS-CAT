#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared1R8C1W8C1G(float *A, float *B, float *C, const int N)
{
// compilador é esperto e aproveita o valor de i, mas faz 1W, 2 R nas outras posições da Shared
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) {
Smem[((threadIdx.x+1)*8)%512] = i;
C[i] = Smem[(threadIdx.x*8)%512];
}
}