#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void coalesced2(float *A, float *C, const int N)
{
int i = (blockIdx.x * blockDim.x + threadIdx.x)*2;

if (i+1 < N) { C[i] = A[i]; C[i+1] = A[i+1];}
}