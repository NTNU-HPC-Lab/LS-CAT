#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(float *in, float *out, int N)
{
// shared memory
__shared__ float smem[BDIM + 2 * RADIUS];

// index to global memory
int idx = blockIdx.x * blockDim.x + threadIdx.x;

while (idx < N)
{

// index to shared memory for stencil calculatioin
int sidx = threadIdx.x + RADIUS;

// Read data from global memory into shared memory
smem[sidx] = in[idx];

// read halo part to shared memory
if (threadIdx.x < RADIUS)
{
smem[sidx - RADIUS] = in[idx - RADIUS];
smem[sidx + BDIM] = in[idx + BDIM];
}

// Synchronize (ensure all the data is available)
__syncthreads();

// Apply the stencil
float tmp = 0.0f;

#pragma unroll
for (int i = 1; i <= RADIUS; i++)
{
tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
}

// Store the result
out[idx] = tmp;

idx += gridDim.x * blockDim.x;
}
}