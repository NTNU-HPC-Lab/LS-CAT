#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPUshared(float *A, float *B, float *C, const int N)
{
__shared__ float smem[512];
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) {
smem[threadIdx.x] += i;
C[i] = A[i] + B[i] + smem[threadIdx.x];
}

}