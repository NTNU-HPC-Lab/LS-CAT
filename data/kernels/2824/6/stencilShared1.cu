#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilShared1(float *src, float *dst, int size, int raio)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
__shared__ float buffer[1024+11];
for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
{
buffer[i] = src[idx+i];
}
idx += raio+1;
if (idx >= size)
return;

__syncthreads();
float out = 0;
#pragma unroll
for(int i = -raio;i < raio; i++)
{
out += buffer[threadIdx.x+raio+i] * const_stencilWeight[i+raio];
}
dst[idx] = out;
}