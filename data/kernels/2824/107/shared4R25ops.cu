#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R25ops(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)
Smem[threadIdx.x] = A[i];
__syncthreads();

float x;
if (i < N) {
x = tan(0.2) *B[i];
x += A[i]/3 + 17*B[i] - A[i]*A[i] + 3*B[i] - 4*A[i]*B[i] + B[i]*B[i]*7;
C[i] = x- 8 +Smem[(threadIdx.x+1)%512]*A[i] + 4*Smem[(threadIdx.x+2)%512]+3*B[i]*Smem[(threadIdx.x+3)%512]+A[i]*Smem[(threadIdx.x+4)%512];
}
}