#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
unsigned int k = i + offset;

if (k + 3 * blockDim.x < n)
{
C[i]                  = A[k]                  + B[k];
C[i + blockDim.x]     = A[k + blockDim.x]     + B[k + blockDim.x];
C[i + 2 * blockDim.x] = A[k + 2 * blockDim.x] + B[k + 2 * blockDim.x];
C[i + 3 * blockDim.x] = A[k + 3 * blockDim.x] + B[k + 3 * blockDim.x];
}
}