#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4Man(float *A, float *B, float *C, const int N)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
float x[4],a[4],b[4],c[4];

a[0] = A[i];
b[0] = B[i];
x[0] = a[0]/7.0;
c[0] = a[0]/3 + 17*b[0] + 3*b[0];
i += blockDim.x * gridDim.x;
a[1] = A[i];
b[1] = B[i];
x[0]*= a[0];
x[1] = a[1]/7.0;
c[1] = a[1]/3 + 17*b[1] + 3*b[1];
x[0]= a[0]*x[0] + x[0]*b[0]*7;
i += blockDim.x * gridDim.x;
a[2] = A[i];
b[2] = B[i];
x[1]*= a[1];
x[2] = a[2]/7.0;
c[2] = a[2]/3 + 17*b[2] + 3*b[2];
x[1]= a[1]*x[1] + x[1]*b[1]*7;
i += blockDim.x * gridDim.x;
if  (i < N) {
a[3] = A[i];
b[3] = B[i];
}
x[2]*= a[2];
x[3] = a[3]/7.0;
c[3] = a[3]/3 + 17*b[3] + 3*b[3];
x[2]= a[2]*x[2] + x[2]*b[2]*7;
x[3]*= a[3];
x[3]= a[3]*x[3] + x[3]*b[3]*7;



i = blockIdx.x * blockDim.x + threadIdx.x;
C[i] += c[0]- x[0];
i += blockDim.x * gridDim.x;
C[i] += c[1]- x[1];
i += blockDim.x * gridDim.x;
C[i] += c[2]- x[2];
i += blockDim.x * gridDim.x;
if  (i < N) C[i] += c[3]- x[3];
}