#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli1(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = poli[idx];

if (idx < N) {
poli[idx] = 3 * x * x - 7 * x + 5;
}
}