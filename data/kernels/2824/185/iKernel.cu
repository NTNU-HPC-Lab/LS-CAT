#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *src, float *dst)
{
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
dst[idx] = src[idx] * 2.0f;
}