#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum10ops(float *A, float *B, float *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i] = A[i] + B[i] - A[i]*A[i] + 3*B[i] - 4*A[i]*B[i] + B[i]*B[i]*7- 8;
}