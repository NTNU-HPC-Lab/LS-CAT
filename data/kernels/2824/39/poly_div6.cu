#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poly_div6(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < N) {
float x = poli[idx];
float y = 1.0/x;
poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+y;
}
}