#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli2(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = poli[idx];

if (idx < N)
poli[idx] = 4 * x * x * x + 3 * x * x - 7 * x + 5;
}