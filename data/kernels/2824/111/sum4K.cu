#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4K(float *A, float *B, float *C, const int N)
{
int j;
int i = blockIdx.x * blockDim.x + threadIdx.x;
float x[4];

#pragma unroll
for (j=0; j < 4; j++)
if  (i < N) {
x[j] = A[i]*A[i];
C[i] += A[i]*3 + 17*B[i] + 3*B[i] - A[i]*x[j] + x[j]*B[i]*7;
i += blockDim.x * gridDim.x;
}
}