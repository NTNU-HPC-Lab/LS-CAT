#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kmeans (short int *input, short int*centroids, int*newcentroids, int *counter, const int n)
{
int Dim = 2;
int i = (blockIdx.x * blockDim.x + threadIdx.x)*Dim;
if ( i < n ) {
// map
int point_d0 = input[i+0];
int point_d1 = input[i+1];
int k0_d0 = point_d0 - centroids[0];
int k0_d1 = point_d1 - centroids[1];
int k1_d0 = point_d0 - centroids[2];
int k1_d1 = point_d1 - centroids[3];
k0_d0 *= k0_d0;
k0_d1 *= k0_d1;
k1_d0 *= k1_d0;
k1_d1 *= k1_d1;
// reduce sum
k0_d0 = k0_d0 + k0_d1;
k1_d0 = k1_d0 + k1_d1;
// reduce min
int k = (k0_d0 < k1_d0 ) ? 0 : 1;
// add current point to new centroids sum
atomicAdd(&(newcentroids[Dim*k]), point_d0);
atomicAdd(&(newcentroids[Dim*k+1]),point_d1);
atomicAdd(&(counter[k]),1);
} // if

}