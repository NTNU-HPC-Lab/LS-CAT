#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void coalesced(float *A, float *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i] = A[i];
}