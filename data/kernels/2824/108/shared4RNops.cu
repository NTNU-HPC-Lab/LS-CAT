#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4RNops(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)
Smem[threadIdx.x] = A[i];
__syncthreads();

float x;
if (i < N) {
x = A[i]/4 + 15*B[i] - A[i]/7.0*A[i] + 4*B[i] + 7*A[i]*A[i] + A[i]*B[i]*7;
x += A[i]/3 + 17*B[i] - A[i]*A[i] + 3*B[i] - 4*A[i]*x + x*B[i]*7;
C[i] = x- 8 +Smem[(threadIdx.x+1)%512]*A[i] + 4*Smem[(threadIdx.x+2)%512]+3*B[i]*Smem[(threadIdx.x+3)%512]+A[i]*Smem[(threadIdx.x+4)%512];
}
}