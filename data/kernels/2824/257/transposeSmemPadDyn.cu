#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPadDyn(float *out, float *in, int nx, int ny)
{
// static shared memory with padding
extern __shared__ float tile[];

// coordinate in original matrix
unsigned int  ix, iy, ti, to;
ix = blockDim.x * blockIdx.x + threadIdx.x;
iy = blockDim.y * blockIdx.y + threadIdx.y;

// linear global memory index for original matrix
ti = iy * nx + ix;

// thread index in transposed block
unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
unsigned int irow    = idx / blockDim.y;
unsigned int icol    = idx % blockDim.y;
unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

// coordinate in transposed matrix
ix = blockDim.y * blockIdx.y + icol;
iy = blockDim.x * blockIdx.x + irow;

// linear global memory index for transposed matrix
to = iy * ny + ix;

// transpose with boundary test
if (ix < nx && iy < ny)
{
// load data from global memory to shared memory
tile[row_idx] = in[ti];

// thread synchronization
__syncthreads();

// store data to global memory from shared memory
out[to] = tile[col_idx];
}
}