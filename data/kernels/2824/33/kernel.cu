#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* polynomial, const size_t N) {
int thread = blockIdx.x * blockDim.x + threadIdx.x;

if (thread < N) {
float x = polynomial[thread];

polynomial[thread] = 3 * x * x - 7 * x + 5;
}
}