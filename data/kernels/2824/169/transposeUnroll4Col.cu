#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

unsigned int ti = iy * nx + ix; // access in rows
unsigned int to = ix * ny + iy; // access in columns

if (ix + 3 * blockDim.x < nx && iy < ny)
{
out[ti]                = in[to];
out[ti +   blockDim.x] = in[to +   blockDim.x * ny];
out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
}
}