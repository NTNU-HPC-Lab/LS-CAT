#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void memcpy( float *dst, float *src )
{

int index = threadIdx.x + 4 * blockIdx.x * blockDim.x;
float a[4];//allocated in registers
for(int i=0;i<4;i++) a[i]=src[index+i*blockDim.x];
for(int i=0;i<4;i++) dst[index+i*blockDim.x]=a[i];
}