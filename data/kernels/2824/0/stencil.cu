#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil(float *src, float *dst, int size, int raio, float *stencilWeight)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx += raio+1;
if (idx >= size)
return;
float out = 0;
#pragma unroll
for(int i = -raio;i < raio; i++)
{
out += src[idx+i] * stencilWeight[i+raio];
}
dst[idx] = out;
}