#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4M(float *A, float *B, float *C, const int N)
{
int j;
int i = blockIdx.x * blockDim.x + threadIdx.x;
float x[4];

#pragma unroll
for (j=0; j < 4; j++)
if  (i < N) {
x[j] = A[i]/7.0*A[i];
C[i] += A[i]/3 + 17*B[i] + 3*B[i] - A[i]*x[j] + x[j]*B[i]*7;
i += blockDim.x * gridDim.x;
}
}