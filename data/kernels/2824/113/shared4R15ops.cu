#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R15ops(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)
Smem[threadIdx.x] = A[i];
__syncthreads();

float x;
if (i < N) {
x = tan(0.2) *B[i];
x += A[i]/3 + 17*B[i];
C[i] = x- 8 +Smem[(threadIdx.x+1)%512]*A[i] + 4*Smem[(threadIdx.x+2)%512]+3*B[i]*Smem[(threadIdx.x+3)%512]+A[i]*Smem[(threadIdx.x+4)%512];
}
}