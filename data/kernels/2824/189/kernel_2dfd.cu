#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
// global to line index
unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

// smem idx for current point
unsigned int stx = threadIdx.x + NPAD;
unsigned int idx  = ix + iStart * nx;

// shared memory for x dimension
__shared__ float line[BDIMX + NPAD2];

// a coefficient related to physical properties
const float alpha = 0.12f;

// register for y value
float yval[9];

for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

// skip for the bottom most y value
int iskip = NPAD * nx;

#pragma unroll 9
for (int iy = iStart; iy < iEnd; iy++)
{
// get yval[8] here
yval[8] = g_u2[idx + iskip];

// read halo part
if(threadIdx.x < NPAD)
{
line[threadIdx.x]  = g_u2[idx - NPAD];
line[stx + BDIMX]    = g_u2[idx + BDIMX];
}

line[stx] = yval[4];
__syncthreads();

// 8rd fd operator
if ( (ix >= NPAD) && (ix < nx - NPAD) )
{
// center point
float tmp = coef[0] * line[stx] * 2.0f;

#pragma unroll
for(int d = 1; d <= 4; d++)
{
tmp += coef[d] * ( line[stx - d] + line[stx + d]);
}

#pragma unroll
for(int d = 1; d <= 4; d++)
{
tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
}

// time dimension
g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
}

#pragma unroll 8
for (int i = 0; i < 8 ; i++)
{
yval[i] = yval[i + 1];
}

// advancd on global idx
idx  += nx;
__syncthreads();
}
}