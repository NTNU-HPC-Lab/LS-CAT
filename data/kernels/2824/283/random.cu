#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void random(float *A, float *B, float *C, const int N)
{
int i = (blockIdx.x * blockDim.x + threadIdx.x);
i = B[i];

if (i < N) C[i] = A[i];
}