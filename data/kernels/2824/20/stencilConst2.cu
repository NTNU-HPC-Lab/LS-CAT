#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilConst2(float *src, float *dst, int size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx += 11;
if (idx >= size)
return;
float out = 0;
#pragma unroll
for(int i = -10;i < 10; i++)
{
out += src[idx+i] * const_stencilWeight[i+10];
}
dst[idx] = out;
}