#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R1W1G(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) {
Smem[threadIdx.x] = i;
C[i] = Smem[(threadIdx.x+1)%512]+Smem[(threadIdx.x+2)%512]+Smem[(threadIdx.x+3)%512]+Smem[(threadIdx.x+4)%512];
}
}