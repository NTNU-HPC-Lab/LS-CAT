#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli_warp(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

float x;
if (idx < N) {
x = poli[idx];
poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))- 1.0f/x + 3.0f/(x*x) + x/5.0f;
}
poli[idx] = x;
}