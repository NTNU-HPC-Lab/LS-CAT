#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R1W1Gs(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

Smem[threadIdx.x] = i;
__syncthreads();

if (i < N) {
C[i] = Smem[(threadIdx.x+1)%512]+Smem[(threadIdx.x+2)%512]+Smem[(threadIdx.x+3)%512]+Smem[(threadIdx.x+4)%512];
}
}