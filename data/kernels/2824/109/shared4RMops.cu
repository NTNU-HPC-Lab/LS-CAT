#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4RMops(float *A, float *B, float *C, const int N)
{
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)
Smem[threadIdx.x] = A[i];
__syncthreads();

float x;
if (i < N) {
x = A[i]/7.0*A[i];
x += A[i]/3 + 17*B[i] + 3*B[i] - A[i]*x + x*B[i]*7;
C[i] = x- 8 +Smem[(threadIdx.x+1)%512]*A[i] + 4*Smem[(threadIdx.x+2)%512]+3*B[i]*Smem[(threadIdx.x+3)%512]+A[i]*Smem[(threadIdx.x+4)%512];
}
}