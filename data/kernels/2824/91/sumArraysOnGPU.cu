#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(double *A, double *B, double *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N) C[i] = A[i] + B[i] + 7*A[i] + 4*B[i]/123.1 - B[i]*A[i] + B[i]*B[i] - 9*B[i]*B[i]*B[i]/0.4 + A[i]/0.2 + B[i]*B[i];
}