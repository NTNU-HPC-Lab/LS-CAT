#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilConst1(float *src, float *dst, int size, int raio)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx += raio+1;
if (idx >= size)
return;
float out = 0;
#pragma unroll
for(int i = -raio;i < raio; i++)
{
out += src[idx+i] * const_stencilWeight[i+raio];
}
dst[idx] = out;
}