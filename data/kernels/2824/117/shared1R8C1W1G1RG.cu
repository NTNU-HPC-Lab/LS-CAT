#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared1R8C1W1G1RG(float *A, float *B, float *C, const int N)
{
// compilador é esperto e aproveita o valor de i, mas faz 1W, 2 R nas outras posições da Shared
__shared__ float Smem[512];

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) {
Smem[(threadIdx.x+1)%512] = A[i];
C[i] = Smem[(threadIdx.x*8)%512];
}
/*if ( blockIdx.x ==  2 && threadIdx.x < 32 ) {
printf("th %d smem %d\n",threadIdx.x,(threadIdx.x*8)%512);
}*/
}