#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void skip_128b(float *A, float *C, const int N)
{
int i = (blockIdx.x * blockDim.x + threadIdx.x)+32*(threadIdx.x%32);

if (i < N) C[i] = A[i];
}