#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli4(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = poli[idx];

if (idx < N)
poli[idx] = 5 + 5 * x + 5 * x * sqrt(x) + 5 * sqrt(x) * x * x + 5 * x *
sqrt(x) * x * x + 5 * x * sqrt(x) * sqrt(x) * x * x;
}