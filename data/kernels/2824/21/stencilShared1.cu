#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilShared1(float *src, float *dst, int size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
__shared__ float buffer[1024+21];
for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
{
buffer[i] = src[idx+i];
}
idx += 11;
if (idx >= size)
return;

__syncthreads();
float out = 0;
#pragma unroll
for(int i = -10;i < 10; i++)
{
out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
}
dst[idx] = out;
}