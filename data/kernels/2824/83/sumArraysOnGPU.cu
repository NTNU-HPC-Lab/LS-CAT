#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i] = 2*A[i] + 3*B[i] - A[i] - 2*B[i];
}