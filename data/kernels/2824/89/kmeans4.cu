#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kmeans4 (short int *input, short int*centroids, int*newcentroids, int *counter, const int n)
{
int Dim = 4;
int i = (blockIdx.x * blockDim.x + threadIdx.x)*Dim;
if ( i < n ) {
// map
int point_d0 = input[i+0];
int point_d1 = input[i+1];
int point_d2 = input[i+2];
int point_d3 = input[i+3];

int k0_d0 = point_d0 - centroids[0];
int k0_d1 = point_d1 - centroids[1];
int k0_d2 = point_d2 - centroids[2];
int k0_d3 = point_d3 - centroids[3];

int k1_d0 = point_d0 - centroids[4];
int k1_d1 = point_d1 - centroids[5];
int k1_d2 = point_d2 - centroids[6];
int k1_d3 = point_d3 - centroids[7];

int k2_d0 = point_d0 - centroids[8];
int k2_d1 = point_d1 - centroids[9];
int k2_d2 = point_d2 - centroids[10];
int k2_d3 = point_d3 - centroids[11];

int k3_d0 = point_d0 - centroids[12];
int k3_d1 = point_d1 - centroids[13];
int k3_d2 = point_d2 - centroids[14];
int k3_d3 = point_d3 - centroids[15];


k0_d0 *= k0_d0;
k0_d1 *= k0_d1;
k0_d2 *= k0_d2;
k0_d3 *= k0_d3;

k1_d0 *= k1_d0;
k1_d1 *= k1_d1;
k1_d2 *= k1_d2;
k1_d3 *= k1_d3;

k2_d0 *= k2_d0;
k2_d1 *= k2_d1;
k2_d2 *= k2_d2;
k2_d3 *= k2_d3;

k3_d0 *= k3_d0;
k3_d1 *= k3_d1;
k3_d2 *= k3_d2;
k3_d3 *= k3_d3;

// reduce sum
k0_d0 = k0_d0 + k0_d1 + k0_d2 + k0_d3;
k1_d0 = k1_d0 + k1_d1 + k1_d2 + k1_d3;
k2_d0 = k2_d0 + k2_d1 + k2_d2 + k2_d3;
k3_d0 = k3_d0 + k3_d1 + k3_d2 + k3_d3;

// reduce min
int k01 = (k0_d0 < k1_d0 ) ? 0 : 1;
int mink01 = (k0_d0 < k1_d0 ) ?  k0_d0: k1_d0 ;
int k23 = (k2_d0 < k3_d0 ) ? 2 : 3;
int mink23 = (k2_d0 < k3_d0 ) ?  k2_d0: k3_d0 ;

int k = (mink01 < mink23) ? k01 : k23;

// add current point to new centroids sum

newcentroids[Dim*k] += point_d0;
newcentroids[Dim*k+1] +=point_d1;
newcentroids[Dim*k+2] += point_d2;
newcentroids[Dim*k+3] += point_d3;
counter[k]++;

} // if

}