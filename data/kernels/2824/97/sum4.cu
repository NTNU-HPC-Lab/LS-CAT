#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4(float *A, float *B, float *C, const int N)
{
int j;
int i = blockIdx.x * blockDim.x + threadIdx.x;

#pragma unroll
for (j=0; j < 4; j++)
if  (i < N) {
C[i] = A[i] + B[i];
i += blockDim.x * gridDim.x;
}
}