#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
int i = threadIdx.x;

if (i < N) C[i] = A[i] + B[i];
}