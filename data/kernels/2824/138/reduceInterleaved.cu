#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check
if(idx >= n) return;

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}