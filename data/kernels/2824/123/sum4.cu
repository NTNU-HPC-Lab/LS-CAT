#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4(float4 *A, float4 *B, float4 *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{ C[i].x = A[i].x + B[i].x;C[i].y = A[i].y + B[i].y;C[i].z = A[i].z + B[i].z;C[i].w = A[i].w + B[i].w;}
}