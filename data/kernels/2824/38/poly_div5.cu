#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poly_div5(float* poli, const int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < N) {
float x = poli[idx];
poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+1.0/x;
}
}