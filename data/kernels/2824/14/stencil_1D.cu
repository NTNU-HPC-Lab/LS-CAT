#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1D(int *in, int *out, int dim){

__shared__ int temp[BLOCKSIZE + 2*RADIUS];

int lindex = threadIdx.x + RADIUS;
int gindex = threadIdx.x + blockDim.x * blockIdx.x;
int stride = gridDim.x * blockDim.x;
int left, right;

// Go through all data
// Step all threads in a block to avoid synchronization problem
while ( gindex < (dim + blockDim.x) ) {

// Read input elements into shared memory
temp[lindex] = 0;
if (gindex < dim)
temp[lindex] = in[gindex];

// Populate halos, set to zero if we are at the boundary
if (threadIdx.x < RADIUS) {

temp[lindex - RADIUS] = 0;
left = gindex - RADIUS;
if (left >= 0)
temp[lindex - RADIUS] = in[left];

temp[lindex + blockDim.x] = 0;
right = gindex + blockDim.x;
if (right < dim)
temp[lindex + blockDim.x] = in[right];
}

// Synchronize threads - make sure all data is available!
__syncthreads();

// Apply the stencil
int result = 0;
for (int offset = -RADIUS; offset <= RADIUS; offset++) {
result += temp[lindex + offset];
}

// Store the result
if (gindex < dim)
out[gindex] = result;

// Update global index and quit if we are done
gindex += stride;

__syncthreads();

}

}