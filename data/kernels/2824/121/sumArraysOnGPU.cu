#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
float d[16];

if (i < N) {
d[threadIdx.x%16]= A[i] + B[i];
C[i] = d[threadIdx.x%8];
}
}