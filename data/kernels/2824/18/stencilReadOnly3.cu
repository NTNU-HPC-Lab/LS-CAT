#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilReadOnly3(float *src, float *dst, int size, float* stencilWeight)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
idx += 11;
if (idx >= size)
return;
float out = 0;
#pragma unroll
for(int i = -10;i < 10; i++)
{
out += src[idx+i] * stencilWeight[i+10];
}
dst[idx] = out;
}