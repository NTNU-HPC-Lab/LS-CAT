#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *A, float *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x*16;

if (i < N) C[i] = A[i];
}