#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
__shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int block = blockIdx.x/(NUM_SCALES+3);
const int scale = blockIdx.x - (NUM_SCALES+3)*block;
const int miny = blockIdx.y*CONVCOL_H;
const int maxy = min(miny + CONVCOL_H, height) - 1;
const int totStart = miny - RADIUS;
const int totEnd = maxy + RADIUS;
const int colStart = block*CONVCOL_W + tx;
const int colEnd = colStart + (height-1)*pitch;
const int sStep = CONVCOL_W*CONVCOL_S;
const int gStep = pitch*CONVCOL_S;
float *kernel = d_Kernel + scale*16;
const int size = pitch*height*scale;
d_Result += size;
d_Data += size;

if (colStart<width) {
float *sdata = data + ty*CONVCOL_W + tx;
int gPos = colStart + (totStart + ty)*pitch;
for (int y = totStart+ty;y<=totEnd;y+=blockDim.y){
if (y<0)
sdata[0] = d_Data[colStart];
else if (y>=height)
sdata[0] = d_Data[colEnd];
else
sdata[0] = d_Data[gPos];
sdata += sStep;
gPos += gStep;
}
}
__syncthreads();
if (colStart<width) {
float *sdata = data + ty*CONVCOL_W + tx;
int gPos = colStart + (miny + ty)*pitch;
for (int y=miny+ty;y<=maxy;y+=blockDim.y) {
d_Result[gPos] =
(sdata[0*CONVCOL_W] + sdata[8*CONVCOL_W])*kernel[0] +
(sdata[1*CONVCOL_W] + sdata[7*CONVCOL_W])*kernel[1] +
(sdata[2*CONVCOL_W] + sdata[6*CONVCOL_W])*kernel[2] +
(sdata[3*CONVCOL_W] + sdata[5*CONVCOL_W])*kernel[3] +
sdata[4*CONVCOL_W]*kernel[4];
sdata += sStep;
gPos += gStep;
}
}
}