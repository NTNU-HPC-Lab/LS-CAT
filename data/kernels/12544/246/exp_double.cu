#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_double(int n,int idx,double *dy,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] =  sin(dy[i]);
}

}