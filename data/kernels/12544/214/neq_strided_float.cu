#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void neq_strided_float(int n, int xOffset,int yOffset,float *dx, float *dy,int incx,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i >= yOffset &&  i % incx == 0 && i % incy == 0)
result[i] = dy[i] != dx[i];
}

}