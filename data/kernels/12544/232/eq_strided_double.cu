#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void eq_strided_double(int n, int xOffset,int yOffset,double *dx,double *dy,int incx,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0)
result[i] = dy[i] == dx[i];
}

}