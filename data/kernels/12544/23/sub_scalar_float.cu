#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"



__global__ void sub_scalar_float(int n,int idx, float dx,float *dy,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx &&  i % incy == 0)
result[i] = dy[i] - dx;
}

}