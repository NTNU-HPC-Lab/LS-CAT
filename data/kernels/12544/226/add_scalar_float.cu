#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_scalar_float(int n, int idx,float dx,float *dy,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i>= idx && i % incy == 0)
result[i] = dy[i] + dx;
}

}