#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_double(int n, double *a, double *b, double *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] * b[i];
}

}