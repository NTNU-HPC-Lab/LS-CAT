#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniform_double(int n,double lower,double upper,double *result) {
int totalThreads = gridDim.x * blockDim.x;
int tid = threadIdx.x;
int i = blockIdx.x * blockDim.x + tid;

for(; i < n; i += totalThreads) {
double u = result[i];
result[i] = u * upper + (1 - u) * lower;
}
}