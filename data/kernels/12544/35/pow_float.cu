#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void pow_float(int n,int idx,float *dy,int incy,float raise,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] =  powf(dy[i],raise);
}

}