#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"



__global__ void rsub_strided_float(int n, int xOffset,int yOffset,float *dx, float *dy,int incx,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i >= yOffset &&  i % incx == 0 && i % incy == 0)
result[i] = dx[i] - dy[i];
}

}