#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void div_scalar_double(int n, int idx,double dx,double *dy,int incy,double * result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] = dy[i] / dx;
}

}