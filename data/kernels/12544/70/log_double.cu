#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void log_double(int n,int idx,double *dy,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] =  log(dy[i]);
}

}