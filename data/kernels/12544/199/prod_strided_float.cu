#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prod_strided_float(int n, int xOffset,float *dx,int incx,float result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i % incx == 0)
result *= dx[i];
}

}