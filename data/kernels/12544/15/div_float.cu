#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_float(int n, float *a, float *b, float *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] / b[i];
}

}