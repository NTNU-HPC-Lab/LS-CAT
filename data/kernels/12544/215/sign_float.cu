#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sign_float(int n,int idx,float *dy,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0) {
float x = dy[i];
result[i] =  (x > 0) - (x < 0);
}
}

}