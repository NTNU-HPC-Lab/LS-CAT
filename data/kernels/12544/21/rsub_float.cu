#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_float(int n, float *a, float *b, float *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = b[i] - a[i];
}

}