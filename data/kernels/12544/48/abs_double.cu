#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void abs_double(int n,int idx,double *dy,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] =  abs(dy[i]);
}

}