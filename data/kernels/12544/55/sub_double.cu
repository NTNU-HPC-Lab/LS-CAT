#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sub_double(int n, double *a, double *b, double *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] - b[i];
}

}