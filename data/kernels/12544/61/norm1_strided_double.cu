#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"


__global__ void norm1_strided_double(int n, int xOffset,double *dx,int incx,double result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i % incx == 0)
result += abs(dx[i]);
}

}