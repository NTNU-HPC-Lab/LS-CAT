#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rsub_double(int n, double *a, double *b, double *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = b[i] - a[i];
}

}