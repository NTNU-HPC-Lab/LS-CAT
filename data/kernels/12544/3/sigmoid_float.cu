#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"


__global__ void sigmoid_float(int n,int idx,float *dy,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] =  1.0 / (1.0 + expf(-dy[i]));
}

}