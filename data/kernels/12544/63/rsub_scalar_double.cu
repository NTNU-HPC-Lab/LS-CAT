#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"



__global__ void rsub_scalar_double(int n, int idx,double dx,double *dy,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] = dx - dy[i];
}

}