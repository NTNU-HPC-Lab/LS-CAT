#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rdiv_strided_float(int n,int xOffset,int yOffset, float *dx,float  *dy,int incx,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i >= yOffset &&  i % incx == 0 && i % incy == 0)
result[i] = dx[i] / dy[i];
}

}