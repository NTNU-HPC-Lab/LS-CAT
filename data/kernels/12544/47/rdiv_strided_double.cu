#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"



__global__ void rdiv_strided_double(int n,int xOffset,int yOffset, double *dx, double *dy,int incx,int incy,double *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= xOffset && i >= yOffset &&  i % incx == 0 && i % incy == 0)
result[i] = dx[i] - dy[i];
}

}