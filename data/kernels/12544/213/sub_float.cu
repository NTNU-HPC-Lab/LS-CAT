#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_float(int n, float *a, float *b, float *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] - b[i];
}

}