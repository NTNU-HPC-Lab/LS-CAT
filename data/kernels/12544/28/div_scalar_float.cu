#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"



__global__ void div_scalar_float(int n,int idx, float dx,float *dy,int incy,float *result) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
if(i >= idx && i % incy == 0)
result[i] = dy[i] / dx;
}

}