#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ShortestPath2(float *Arr1,float *Arr2,float *recv,int N,int rows, int k,int rank,int owner){

int col=blockIdx.x * blockDim.x + threadIdx.x;
int row=blockIdx.y * blockDim.y + threadIdx.y;

int index=row*N+col;
int index_ik = row*N+k;


if(Arr1[index]>(Arr1[index_ik]+recv[col])){
Arr2[index]=Arr1[index_ik]+recv[col];
}
__syncthreads();

}