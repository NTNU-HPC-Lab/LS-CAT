#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length
__global__ void ShortestPath3(float *Arr1,float *Arr2,int N){//Arr1 input array,Holds weights
//Arr2 output array
unsigned int k;
int row=blockIdx.x;
int col=threadIdx.x;

if(row >= N || col >= N) return;

int index=row*N+col;					//Index for Elements of global memory array
extern __shared__ float sArr[];			//Dynamic share memory allocation in Array
int stride=N/CELLS_PER_THREAD;			// Stride for each index of arrays (both global and shared)

for(stride=0;stride<N;stride+=N/CELLS_PER_THREAD){
if(threadIdx.x+stride>N) break;						//Copy elements of same ROW in shared memory (SHARED memory indexes = ThreadIdx.x + stride)
sArr[threadIdx.x+stride]=Arr1[index+stride];		//"jump" to the next index according to stride .Write to SM
}
__syncthreads();										//Wait all threads in current block


for(stride=0;stride<N;stride+=N/CELLS_PER_THREAD){

for(k=0;k<N;k++){
if(k==threadIdx.x+stride)continue;
if(sArr[threadIdx.x+stride]>(sArr[k]+Arr1[k*N+col+stride])){		//Return
Arr2[index+stride]=(sArr[k]+Arr1[k*N+col+stride]);		//If Condition true,write to shared memory
//
}
}
//						//Copy results to global memory and return to HOST

}
}