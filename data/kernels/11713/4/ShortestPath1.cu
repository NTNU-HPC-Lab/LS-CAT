#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ShortestPath1(float *Arr1,float *Arr2,int N,int rows, int rank){     //rowNum is number of rows for each process (full assigned to process)
//Arr1 input array,Holds  of (u,v)	//Arr2 output array
int k;
int col=blockIdx.x * blockDim.x + threadIdx.x;
int row=blockIdx.y * blockDim.y + threadIdx.y;

int offset=rows*rank;

int index=row*N+col;
int index_ik,index_kj;

Arr2[index]=Arr1[index];
for(k=rank*rows; k<((rank+1)*rows); k++){

index_ik = row*N+k;
index_kj = (k-offset)*N+col;

if(Arr1[index]>(Arr1[index_ik]+Arr1[index_kj])){
Arr2[index]=Arr1[index_ik]+Arr1[index_kj];
}
__syncthreads();
}
}