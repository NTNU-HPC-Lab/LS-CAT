#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length
__global__ void ShortestPath2(float *Arr1,float *Arr2,int N){   //Arr1 input array,Holds weights
//Arr2 output array
unsigned int k;

int row=blockIdx.x;
int col=threadIdx.x;
if(row >= N || col >= N) return;

int index=row*N+col;						//Index for Elements of global memory array
extern __shared__ float sArr[];				//Dynamic share memory allocation in Array
Arr2[index]=Arr1[index];
sArr[threadIdx.x]=Arr1[index];				//Copy elements of same ROW in shared memory (SHARED memory indexes = ThreadIdx.x )
__syncthreads();							//Wait threads in block to finish

for(k=0;k<N;k++){
if(k==threadIdx.x) continue;						//If out of bounds , just current loop iteration
if(sArr[threadIdx.x]>(sArr[k]+Arr1[k*N+col])){		//If condition true write in shared memory new value
Arr2[index]=(sArr[k]+Arr1[k*N+col]);
}

}
}