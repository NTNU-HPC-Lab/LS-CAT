#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length
__global__ void ShortestPath1(float *Arr1,float *Arr2,int N){
//Arr1 input array,Holds  of (u,v)
//Arr2 output array
int k;
int col=blockIdx.x * blockDim.x + threadIdx.x;
int row=blockIdx.y * blockDim.y + threadIdx.y;
int index=row*N+col;

if((row<N)&&(col<N)){
Arr2[index]=Arr1[index];

for(k=0;k<N;k++){

if(Arr1[index]>(Arr1[row*N+k]+Arr1[N*k+col])){
Arr2[index]=Arr1[row*N+k]+Arr1[N*k+col];
//	printf("ENTERED %f \n",Arr2[index]);
}
}
}

}