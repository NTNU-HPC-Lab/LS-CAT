#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Example of how to use the mxGPUArray API in a MEX file.  This example shows
* how to write a MEX function that takes a gpuArray input and returns a
* gpuArray output, e.g. B=mexFunction(A).
*
* Copyright 2012 The MathWorks, Inc.
*/


#define DIVUP(m,n)		((m)/(n)+((m)%(n)>0))
int const threadsPerBlock = (sizeof(unsigned long long) * 8);

/*
* Device code
*/
__device__ inline float devIoU(float const * const a, float const * const b)
{
float left = max(a[0], b[0]), right = min(a[2], b[2]);
float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
float interS = width * height;
float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
return interS / (Sa + Sb - interS);
}
__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thres, const float *dev_boxes, unsigned long long *dev_mask)
{
const int row_start = blockIdx.y, col_start = blockIdx.x;
const int row_size = min(n_boxes - row_start * threadsPerBlock, threadsPerBlock), col_size = min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

//if (row_start > col_start) return;

__shared__ float block_boxes[threadsPerBlock * 5];
if (threadIdx.x < col_size)
{
block_boxes[threadIdx.x * 5 + 0] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
block_boxes[threadIdx.x * 5 + 1] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
block_boxes[threadIdx.x * 5 + 2] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
block_boxes[threadIdx.x * 5 + 3] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
block_boxes[threadIdx.x * 5 + 4] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
}
__syncthreads();

if (threadIdx.x < row_size)
{
const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
const float *cur_box = dev_boxes + cur_box_idx * 5;
int i = 0;
unsigned long long t = 0;
int start = 0;
if (row_start == col_start) start = threadIdx.x + 1;
for (i = start; i < col_size; i++)
{
if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thres)
{
t |= 1ULL << i;
}
}
const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
dev_mask[cur_box_idx * col_blocks + col_start] = t;
}
}