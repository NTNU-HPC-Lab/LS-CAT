#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 30000				//Length and width of inner grid in threads
#define DIM (SIZE + 2)			//Length and width of the entire grid in threads
#define GRID_SIZE 1500 			//Length and width of inner grid in blocks
#define BLOCK_SIZE 20 			//Length and width of block in threads
#define MEM_SIZE (sizeof(float) * DIM * DIM)
#define TIME_STEPS 1
#define PINNED 0

void fillGrid(float* grid);

__global__ void computeGrid(float* read, float* write) {
//Retrieve the thread's position in the grid
//The position is offset by 1 in the x and y directions to remove boundary checks
int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

//Writes the sum of the neighbors to the cell
write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
}