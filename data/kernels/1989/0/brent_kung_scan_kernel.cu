#include "hip/hip_runtime.h"
#include "includes.h"


constexpr const int SECTION_SIZE = 2048;
constexpr const int MAX_SECTIONS = 1024;

__device__ void brent_kung_scan_(float *X, float *Y, int InputSize) {
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int bdx = blockDim.x;

__shared__ float XY[SECTION_SIZE];
int i = 2 * bx * bdx + tx;
if (i < InputSize)
XY[tx] = X[i];
if (i + bdx < InputSize)
XY[tx + bdx] = X[i + bdx];
for (unsigned int stride = 1; stride <= bdx; stride *= 2) {
__syncthreads();
int index = (tx + 1) * 2 * stride - 1;
if (index < SECTION_SIZE) {
XY[index] += XY[index - stride];
}
}
for (int stride = SECTION_SIZE / 4; stride > 0; stride /= 2) {
__syncthreads();
int index = (tx + 1) * stride * 2 - 1;
if (index + stride < SECTION_SIZE) {
XY[index + stride] += XY[index];
}
}
__syncthreads();
if (i < InputSize)
Y[i] = XY[tx];
if (i + bdx < InputSize)
Y[i + bdx] = XY[tx + bdx];
}
__global__ void brent_kung_scan_kernel(float *X, float *Y, int InputSize) {
brent_kung_scan_(X, Y, InputSize);
}