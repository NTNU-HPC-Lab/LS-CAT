#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_privatized_kernel(unsigned char *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {
const int bx = blockIdx.x;
const int bdx = blockDim.x;
const int tx = threadIdx.x;
const int gdx = gridDim.x;
unsigned int tid = bx * bdx + tx;

extern __shared__ unsigned int histo_s[]; // size is 3rd arg in <<< >>> of kernel
for (unsigned int bin_idx = tx; bin_idx < num_bins; bin_idx += bdx) {
histo_s[bin_idx] = 0u;
}
__syncthreads();

const int bin_size = (num_elements - 1) / num_bins + 1;
for (unsigned int i = tid; i < num_elements; i += bdx * gdx) {
int c = input[i] - 'a';
if (c >= 0 && c < 26)
atomicAdd(&(histo_s[c / bin_size]), 1);
}
__syncthreads();

for (unsigned int bin_idx = tx; bin_idx < num_bins; bin_idx += bdx) {
atomicAdd(&(bins[bin_idx]), histo_s[bin_idx]);
}
}