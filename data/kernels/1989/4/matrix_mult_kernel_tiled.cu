#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_mult_kernel_tiled(int *d_m, int *d_n, int *d_p, int m, int n, int k) {
/*
* [m][k] @ [k][n] = [m][n]
*/
__shared__ int ds_m[TILE_WIDTH][TILE_WIDTH]; // ds: device shared memory
__shared__ int ds_n[TILE_WIDTH][TILE_WIDTH];

int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x;
int by = blockIdx.y;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

int pvalue = 0;

for (int i = 0; i < ceil(k / (float)TILE_WIDTH); ++i) {
// thread collaborative loading into shared memory
if (row < m && (i * TILE_WIDTH + tx) < k)
ds_m[ty][tx] = d_m[row * k + i * TILE_WIDTH + tx]; // coalesced
else
ds_m[ty][tx] = 0;
if (col < n && (i * TILE_WIDTH + ty) < k)
ds_n[ty][tx] = d_n[(i * TILE_WIDTH + ty) * n + col]; // coalesced
else
ds_n[ty][tx] = 0;

__syncthreads();

for (int j = 0; j < TILE_WIDTH; j++)
pvalue += ds_m[ty][j] * ds_n[j][tx];
__syncthreads();
}

if (row < m && col < n)
d_p[row * n + col] = pvalue;
}