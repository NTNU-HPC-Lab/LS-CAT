#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_mult_kernel(int *a, int *b, int *c, int m, int n, int k) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if (col < n && row < m) {
for (int i = 0; i < k; i++) {
sum += a[row * k + i] * b[i * n + col];
}
c[row * n + col] = sum;
}
}