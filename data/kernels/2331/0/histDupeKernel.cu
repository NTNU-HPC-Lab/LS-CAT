#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histDupeKernel(const float* data1, const float* data2, const float* confidence1, const float* confidence2, int* ids1, int* ids2, int* results_id1, int* results_id2, float* results_similarity, int* result_count, const int N1, const int N2, const int max_results) {

const unsigned int thread = threadIdx.x; // Thread index within block
const unsigned int block = blockIdx.x; // Block index
const unsigned int block_size = blockDim.x; // Size of each block

const unsigned int block_start = block_size * block; // Index of the start of the block
const unsigned int index = block_start + thread; // Index of this thread

//__shared__ float conf[64]; // Shared array of confidence values for all histograms owned by this block
//conf[thread] = confidence1[index]; // Coalesced read of confidence values
float conf = confidence1[index];
int id = ids1[index];

__shared__ float hists[128 * 64]; // Shared array of all histograms owned by this block
for (unsigned int i = 0; i < 64; i++) {
hists[i * 128 + thread] = data1[(block_start + i) * 128 + thread]; // Coalesced read of first half of histogram
hists[i * 128 + thread + 64] = data1[(block_start + i) * 128 + 64 + thread]; // Coalesced read of second half of histogram
}

__shared__ float other[128]; // Histogram to compare all owned histograms against parallely
for (unsigned int i = 0; i < N2 && *result_count < max_results; i++) {

other[thread] = data2[i * 128 + thread]; // Coalesced read of first half of other histogram
other[thread + 64] = data2[i * 128 + thread + 64]; // Second half

__syncthreads(); // Ensure all values read

if (index < N1) {
float d = 0;
for (unsigned int k = 0; k < 128; k++) { // Compute sum of distances between thread-owned histogram and shared histogram
d += fabsf(hists[thread * 128 + k] - other[k]);
}
d = 1 - (d / 8); // Massage the difference into a nice % similarity number, between 0 and 1

int other_id = ids2[i];

if (other_id != id && d > fmaxf(conf, confidence2[i])) { // Don't compare against self, only compare using highest confidence
int result_index = atomicAdd(result_count, 1); // Increment result count by one atomically (returns value before increment)
if (result_index < max_results) {
// Store resulting pair
results_similarity[result_index] = d;
results_id1[result_index] = id;
results_id2[result_index] = other_id;
}
}
}

__syncthreads(); // Ensure all threads have finished before looping and reading new shared histogram
}

}