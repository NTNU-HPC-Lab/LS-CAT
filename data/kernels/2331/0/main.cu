#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "histDupeKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *data1 = NULL;
hipMalloc(&data1, XSIZE*YSIZE);
const float *data2 = NULL;
hipMalloc(&data2, XSIZE*YSIZE);
const float *confidence1 = NULL;
hipMalloc(&confidence1, XSIZE*YSIZE);
const float *confidence2 = NULL;
hipMalloc(&confidence2, XSIZE*YSIZE);
int *ids1 = NULL;
hipMalloc(&ids1, XSIZE*YSIZE);
int *ids2 = NULL;
hipMalloc(&ids2, XSIZE*YSIZE);
int *results_id1 = NULL;
hipMalloc(&results_id1, XSIZE*YSIZE);
int *results_id2 = NULL;
hipMalloc(&results_id2, XSIZE*YSIZE);
float *results_similarity = NULL;
hipMalloc(&results_similarity, XSIZE*YSIZE);
int *result_count = NULL;
hipMalloc(&result_count, XSIZE*YSIZE);
const int N1 = 1;
const int N2 = 1;
const int max_results = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
histDupeKernel<<<gridBlock,threadBlock>>>(data1,data2,confidence1,confidence2,ids1,ids2,results_id1,results_id2,results_similarity,result_count,N1,N2,max_results);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
histDupeKernel<<<gridBlock,threadBlock>>>(data1,data2,confidence1,confidence2,ids1,ids2,results_id1,results_id2,results_similarity,result_count,N1,N2,max_results);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
histDupeKernel<<<gridBlock,threadBlock>>>(data1,data2,confidence1,confidence2,ids1,ids2,results_id1,results_id2,results_similarity,result_count,N1,N2,max_results);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}