#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

// don't forget to compile with "nvcc -ptx cudaKernel.cu -o cudaKernel.ptx
// And to move the ptx file in the resources !
__global__ void add(int n, float* a, float* b, float* sum) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride)
sum[i] = a[i] + b[i];
}