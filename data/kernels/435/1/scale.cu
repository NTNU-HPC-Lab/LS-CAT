#include "hip/hip_runtime.h"
#include "includes.h"
/**
* C file for parallel QR factorization program usign CUDA
* See header for more infos.
*
* 2016 Marco Tieghi - marco01.tieghi@student.unife.it
*
*/



#define THREADS_PER_BLOCK 512   //I'll use 512 threads for each block (as required in the assignment)

__global__ void scale(double *d, int m, int ld, double *s) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < m) {
d[idx*ld] = d[idx*ld] / sqrt(*s);    //Applying scale
}
}