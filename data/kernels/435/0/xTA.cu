#include "hip/hip_runtime.h"
#include "includes.h"
/**
* C file for parallel QR factorization program usign CUDA
* See header for more infos.
*
* 2016 Marco Tieghi - marco01.tieghi@student.unife.it
*
*/



#define THREADS_PER_BLOCK 512   //I'll use 512 threads for each block (as required in the assignment)

__global__ void xTA (double *y, int k, double*A, int m, int lda, double *x, int ldx) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
double s;   //It memorizes the sum

if (idx < k) {
for (int ii = 0; ii < m; ii++) {    //Moving through rows
s += x[ii * ldx] * A[idx + ii*lda];
}
y[idx] = s;  //Adding the sum to result vector
}
}