#include "hip/hip_runtime.h"
#include "includes.h"
/**
* C file for parallel QR factorization program usign CUDA
* See header for more infos.
*
* 2016 Marco Tieghi - marco01.tieghi@student.unife.it
*
*/



#define THREADS_PER_BLOCK 512   //I'll use 512 threads for each block (as required in the assignment)

__global__ void r1_update(double *A, int m, int n, int lda, double *col, int ldc, double *row) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;

//A(:,ii+1:n−1)=A(:,ii+1:n−1)−A(:,ii)*R(ii,ii+1:n−1)
if (idx < m && idy < m) {
for (int ii=0; ii < n-1; ii++) {
A[idx*lda + ii+1] = A[idx*lda + ii+1] - col[idy*ldc] * row[ii+1];
}
}
}