#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AssembleArrayOfNoticedChannels ( const int nmbrOfChnnls, const float lwrNtcdEnrg, const float hghrNtcdEnrg, const float *lwrChnnlBndrs, const float *hghrChnnlBndrs, const float *gdQltChnnls, float *ntcdChnnls ) {
int c = threadIdx.x + blockDim.x * blockIdx.x;
if ( c < nmbrOfChnnls ) {
ntcdChnnls[c] = ( lwrChnnlBndrs[c] > lwrNtcdEnrg ) * ( hghrChnnlBndrs[c] < hghrNtcdEnrg ) * ( 1 - gdQltChnnls[c] );
}
}