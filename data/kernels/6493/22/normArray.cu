#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normArray ( const int n, float *a ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
float c = a[0];
if ( i < n ) {
a[i] = a[i] / c;
}
}