#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnQ1 ( const int dim, const int n, const float *p1, const float *p0, const float *s1, const float *s0, const float *zr, float *q ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
if ( p1[i] == INF || - 0.5 * ( s1[i] + p1[i] - s0[i] - p0[i] ) < -10. ) {
q[i] = 0.0;
} else if ( - 0.5 * ( s1[i] + p1[i] - s0[i] - p0[i] ) > 10. ) {
q[i] = 1.E10;
} else {
q[i] = expf ( - 0.5 * ( s1[i] + p1[i] - s0[i] - p0[i] ) ) * powf ( zr[i], dim - 1 );
}
}
}