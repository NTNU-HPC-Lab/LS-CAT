#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleArray ( const int n, const float c, float *a ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
a[i] = c * a[i];
}
}