#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortIndexKde ( const int d, const int n, const float *a, const float *b, float *sa, float *sb ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int ij = i + j * d;
int mewj, il;
float mewa, mewb;
if ( i < d && j < n ) {
mewj = j;
mewa = a[ij];
mewb = b[ij];
for ( int l = 0; l < n; l++ ) {
il = i + l * d;
if ( l > j ) {
mewj += ( a[il] > mewa ) * ( l - mewj );
mewa += ( a[il] > mewa ) * ( a[il] - mewa );
mewb += ( a[il] > mewa ) * ( b[il] - mewb );
} else if ( l < j ) {
mewj += ( a[il] < mewa ) * ( l - mewj );
mewa += ( a[il] < mewa ) * ( a[il] - mewa );
mewb += ( a[il] < mewa ) * ( b[il] - mewb );
}
}
sa[ij] = mewa;
sb[ij] = mewb;
}
}