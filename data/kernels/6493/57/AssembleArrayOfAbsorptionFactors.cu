#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AssembleArrayOfAbsorptionFactors ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int nmbrOfElmnts, const float *crssctns, const float *abndncs, const int *atmcNmbrs, const float *wlkrs, float *absrptnFctrs ) {
int enIndx = threadIdx.x + blockDim.x * blockIdx.x;
int wlIndx = threadIdx.y + blockDim.y * blockIdx.y;
int ttIndx = enIndx + wlIndx * nmbrOfEnrgChnnls;
int elIndx, effElIndx, crIndx, prIndx;
float xsctn, clmn, nh;
if ( enIndx < nmbrOfEnrgChnnls && wlIndx < nmbrOfWlkrs ) {
if ( NHINDX == NPRS-1 ) {
elIndx = 0;
prIndx = elIndx + NHINDX;
crIndx = elIndx + enIndx * nmbrOfElmnts;
effElIndx = atmcNmbrs[elIndx] - 1;
nh = wlkrs[prIndx+wlIndx*NPRS] * 1.E22;
clmn = abndncs[effElIndx];
xsctn = clmn * crssctns[crIndx];
elIndx = 1;
while ( elIndx < nmbrOfElmnts ) {
prIndx = elIndx + NHINDX;
crIndx = elIndx + enIndx * nmbrOfElmnts;
effElIndx = atmcNmbrs[elIndx] - 1;
clmn = abndncs[effElIndx]; // * powf ( 10, wlkrs[wlIndx].par[prIndx] );
xsctn = xsctn + clmn * crssctns[crIndx];
elIndx += 1;
}
absrptnFctrs[ttIndx] = expf ( - nh * xsctn );
} else if ( NHINDX == NPRS ) {
absrptnFctrs[ttIndx] = 1;
}
}
}