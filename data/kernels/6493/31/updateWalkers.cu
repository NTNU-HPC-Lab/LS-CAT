#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateWalkers ( const int dim, const int nwl, const float *xx1, const float *q, const float *r, float *xx0 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
//if ( q[j] > r[j] ) {
xx0[t] = ( q[j] > r[j] ) * xx1[t] + ( q[j] <= r[j] ) * xx0[t];
//}
}
}