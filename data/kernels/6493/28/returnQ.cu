#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnQ ( const int dim, const int n, const float *s1, const float *s0, const float *zr, float *q ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
q[i] = expf ( - 0.5 * ( s1[i] - s0[i] ) ) * powf ( zr[i], dim - 1 );
}
}