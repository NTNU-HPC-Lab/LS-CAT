#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addWalkers ( const int dim, const int nwl, const float *xx0, const float *xxW, float *xx1 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xx1[t] = xx0[t] + xxW[t];
}
}