#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void powWalkers ( const int n, const float c, const float *a, float *d ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
d[i] = powf ( a[i], c );
}
}