#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnQM ( const int dim, const int n, const float *s1, const float *s0, float *q ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
q[i] = expf ( - 0.5 * ( s1[i] - s0[i] ) );
}
}