#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void metropolisPoposal2 ( const int dim, const int nwl, const int isb, const float *xx, const float *rr, float *xx1 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xx1[t] = xx[t] + ( i == isb ) * rr[j];
}
}