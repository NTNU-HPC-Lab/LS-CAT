#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setChiAtLast ( const int dim, const int nwl, const float *lst, float *stt ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
stt[i] = lst[dim+2+i*(dim+1+1+1+1)];
}
}