#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortMatrix ( const int nd, const float *a, float *sm ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int ij = i + j * nd;
if ( i < nd && j < nd ) {
sm[ij] = ( a[i] > a[j] );
}
}