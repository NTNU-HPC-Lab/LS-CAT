#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeAtRandom ( const int dim, const int nwl, const float dlt, const float *x0, const float *stn, float *xx ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xx[t] = x0[i] + dlt * stn[t];
}
}