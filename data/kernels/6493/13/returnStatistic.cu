#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnStatistic ( const int dim, const int nwl, const float *xx, float *s ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
s[t] = powf ( xx[t], 2. );
}
}