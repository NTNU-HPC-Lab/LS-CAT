#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateStatistic ( const int nwl, const float *stt1, const float *q, const float *r, float *stt0 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
stt0[i] = ( q[i] > r[i] ) * stt1[i] + ( q[i] < r[i] ) * stt0[i];
}
}