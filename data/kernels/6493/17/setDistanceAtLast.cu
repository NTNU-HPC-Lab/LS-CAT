#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setDistanceAtLast ( const int dim, const int nwl, const float *lst, float *didi ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
didi[i] = lst[dim+i*(dim+1+1+1+1)];
}
}