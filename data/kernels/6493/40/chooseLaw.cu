#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chooseLaw ( const int nwl, const int *kex, const float *didi11, const float *didi12, const float *didi13, float *didi1 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
didi1[i] = ( kex[i] == 0 ) * didi11[i] + ( kex[i] == 1 ) * didi12[i] + ( kex[i] == 2 ) * didi13[i];
}
}