#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chooseSample ( const int nDB, const int si, const float *EE, float *EBV ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nDB ) {
EBV[i] = EE[i+si*nDB];
}
}