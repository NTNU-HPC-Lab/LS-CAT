#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constantArray ( const int n, const float c, float *a ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
a[i] = c;
}
}