#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chainFunction ( const int dim, const int nwl, const int nst, const int ipr, const float *smpls, float *chnFnctn ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * nwl;
if ( i < nwl && j < nst ) {
chnFnctn[t] = smpls[ipr+t*dim];
}
}