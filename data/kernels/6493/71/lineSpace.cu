#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lineSpace ( const int d, const int n, const float *l, const float *h, float *b ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
float delta;
if ( i < d && j < n ) {
delta = ( h[i] - l[i] ) / ( n - 1 );
b[i+j*d] = l[i] + j * delta;
}
}