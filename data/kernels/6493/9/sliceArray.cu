#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sliceArray ( const int n, const int indx, const float *ss, float *zz ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
zz[i] = ss[i+indx];
}
}