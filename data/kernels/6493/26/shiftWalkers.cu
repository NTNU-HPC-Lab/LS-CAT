#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shiftWalkers ( const int dim, const int nwl, const float *xx, const float *x, float *yy ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
yy[t] = xx[t] - x[i];
}
}