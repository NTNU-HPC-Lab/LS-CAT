#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saveTheWhalesXX ( const int d0, const int d1, const int i2, float *xxx, const int d3, const int d4, const float *xx ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
if ( i < d3 && j < d4 ) {
xxx[i+j*d0+i2*d0*d1] = xx[i+j*d3];
}
}