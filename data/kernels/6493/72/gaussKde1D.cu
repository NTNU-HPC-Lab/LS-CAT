#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaussKde1D ( const int dim, const int nd, const int nb, const int Indx, const float *hh, const float *a, const float *b, float *pdf ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int ij = i + j * nb;
float h;
if ( i < nb && j < nd ) {
h = hh[Indx];
pdf[ij] = expf ( - powf ( a[Indx+j*dim] - b[Indx+i*dim], 2. ) / 2. / powf ( h, 2 ) ) / h / powf ( 2 * PI, 0.5 );
}
}