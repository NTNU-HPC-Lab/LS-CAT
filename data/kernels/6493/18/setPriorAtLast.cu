#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setPriorAtLast ( const int dim, const int nwl, const float *lst, float *prr ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
prr[i] = lst[dim+3+i*(dim+1+1+1+1)];
}
}