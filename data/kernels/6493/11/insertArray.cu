#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void insertArray ( const int n, const int indx, const float *ss, float *zz ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
zz[indx+i] = ss[i];
}
}