#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayOfPriors ( const int dim, const int nwl, const float *cn, const float *xx, float *pr ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
float sum = 0.;
if ( i < nwl ) {
pr[i] = ( cn[i] == dim ) * sum + ( cn[i] < dim ) * INF;
}
}