#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleWalkers ( const int n, const float c, const float *a, float *d ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
d[i] = c * a[i];
}
}