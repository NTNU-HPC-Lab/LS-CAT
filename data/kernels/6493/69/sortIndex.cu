#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortIndex ( const int d, const int n, const float *a, int *si, float *sa ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int ij = i + j * d;
int mewj, il;
float mewa;
if ( i < d && j < n ) {
mewj = j;
mewa = a[ij];
for ( int l = 0; l < n; l++ ) {
il = i + l * d;
if ( l > j ) {
mewj += ( a[il] < mewa ) * ( l - mewj );
mewa += ( a[il] < mewa ) * ( a[il] - mewa );
} else if ( l < j ) {
mewj += ( a[il] > mewa ) * ( l - mewj );
mewa += ( a[il] > mewa ) * ( a[il] - mewa );
}
}
si[ij] = mewj;
sa[ij] = mewa;
}
}