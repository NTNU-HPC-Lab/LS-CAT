#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapRandomNumbers ( const int nwl, const int ist, const int isb, const float *r, float *zr, int *kr, float *ru, int *kex ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int rr;
if ( i < nwl ) {
rr = i + 0 * nwl + isb * 4 * nwl + ist * 4 * 2 * nwl;
zr[i] = 1. / ACONST * powf ( r[rr] * ( ACONST - 1 ) + 1, 2. );
rr = i + 1 * nwl + isb * 4 * nwl + ist * 4 * 2 * nwl;
kr[i] = ( int ) truncf ( r[rr] * ( nwl - 1 + 0.999999 ) );
rr = i + 2 * nwl + isb * 4 * nwl + ist * 4 * 2 * nwl;
ru[i] = r[rr];
rr = i + 3 * nwl + isb * 4 * nwl + ist * 4 * 2 * nwl;
kex[i] = ( int ) truncf ( r[rr] * ( 5 - 1 + 0.999999 ) );
}
}