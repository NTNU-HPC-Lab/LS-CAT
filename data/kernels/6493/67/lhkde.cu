#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lhkde ( const int n, const float *a, const float *b, float *l, float *h  ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
l[i] = a[i] - 3 * b[i];
h[i] = a[i] + 3 * b[i];
}
}