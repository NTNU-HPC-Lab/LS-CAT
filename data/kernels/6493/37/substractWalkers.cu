#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void substractWalkers ( const int dim, const int nwl, const float *xx0, const float *xxCP, float *xx1 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xx1[t] = xx0[t] - xxCP[t];
}
}