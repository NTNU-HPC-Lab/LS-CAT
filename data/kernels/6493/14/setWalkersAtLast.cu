#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setWalkersAtLast ( const int dim, const int nwl, const float *lst, float *xx ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xx[t] = lst[i+j*(dim+1+1+1+1)];
}
}