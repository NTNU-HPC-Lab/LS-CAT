#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayOf2DConditions ( const int dim, const int nwl, const float *bn, const float *xx, float *cc ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
cc[t] = ( bn[0+i*2] < xx[t] ) * ( xx[t] < bn[1+i*2] );
}
}