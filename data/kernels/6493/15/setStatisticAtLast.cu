#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setStatisticAtLast ( const int dim, const int nwl, const float *lst, float *stt ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
stt[i] = lst[dim+1+i*(dim+1+1+1+1)];
}
}