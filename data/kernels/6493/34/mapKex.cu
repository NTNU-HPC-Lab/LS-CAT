#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapKex ( const int nwl, const float *r, int *kex ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
kex[i] = ( int ) truncf ( r[i] * ( 3 - 1 + 0.999999 ) );
}
}