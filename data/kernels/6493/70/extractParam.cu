#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extractParam ( const int d, const int n, const int Indx, const float *s, float *a ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
a[i] = s[Indx+i*d];
}
}