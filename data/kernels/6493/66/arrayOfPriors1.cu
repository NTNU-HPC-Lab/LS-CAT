#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayOfPriors1 ( const int dim, const int nwl, const float *cn, const float *nhMd, const float *nhSg, const float *xx, float *pr ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
float sum; //, theta, kk;
if ( i < nwl ) {
//theta = powf ( nhSg[i], 2 ) / nhMd[i];
//kk = nhMd[i] / theta;
//sum = ( kk - 1 ) * logf ( xx[NHINDX+i*nwl] ) - xx[NHINDX+i*nwl] / theta;
sum = 0; //powf ( ( xx[NHINDX+i*nwl] - nhMd[i] ) / nhSg[i], 2 );
pr[i] = ( cn[i] == dim ) * sum + ( cn[i] < dim ) * INF;
}
}