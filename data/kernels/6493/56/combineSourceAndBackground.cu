#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void combineSourceAndBackground ( const int nwl, const int n, const float scale, float *src, const float *bkg ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
if ( i < n && j < nwl ) {
src[i+j*n] = src[i+j*n] + scale * bkg[i+j*n];
}
}