#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chooseDistance ( const int nwl, const int *kex, const float *didi11, float *didi1 ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < nwl ) {
didi1[i] = didi11[i+kex[i]*nwl];
}
}