#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TestpermuteWalkers ( const int dim, const int nwl, const int *kr, const float *xxC, float *xxCP ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = threadIdx.y + blockDim.y * blockIdx.y;
int t = i + j * dim;
if ( i < dim && j < nwl ) {
xxCP[t] = xxC[t];
}
}