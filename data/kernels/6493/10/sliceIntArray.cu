#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sliceIntArray ( const int n, const int indx, const int *ss, int *zz ) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
if ( i < n ) {
zz[i] = ss[i+indx];
}
}