#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int oW = iW - kL + 1;
int x_offset = (i/oW)*iW + i%oW;
int w_offset = (i/(oW*iH))*kL;

for (int k = 0; k < kL; k++) {
y[i] += w[w_offset + k]*x[x_offset + k];
}
}
}