#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int iW = oW + kL - 1;
int col = i%iW;
int dy_offset = (i/iW)*oW + i%iW;
int w_offset = (i/(iW*oH))*kL;

int k_begin = max(0, col-oW+1);
int k_end = min(kL, col+1);

dx[i] = 0.0f;
for (int k = k_begin; k < k_end; k++) {
dx[i] += w[w_offset + k]*dy[dy_offset - k];
}
}
}