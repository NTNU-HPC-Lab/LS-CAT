#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_vertical_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int dy_offset = (i/kL)*oH*oW;
int x_offset = (i/kL)*oH*oW + (i%kL)*oW;

for (int k = 0; k < oH*oW; k++) {
dw[i] += dy[dy_offset + k]*x[x_offset + k];
}
}
}