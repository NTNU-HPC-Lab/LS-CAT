#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef TIME
#define COMM 1
#elif NOTIME
#define COMM 0
#endif

#define MASK_WIDTH 5
#define TILE_WIDTH 32
#define GPU 1
#define COMMENT "skeletization_GPU"
#define RGB_COMPONENT_COLOR 255


typedef struct {
unsigned char red, green, blue;
} PPMPixel;

typedef struct {
int x, y;
PPMPixel *data;
} PPMImage;

typedef struct {
int x, y;
} Par;

double time_total;
__global__ void Condition(int *GrayScale_, int *d_changing1, int *cont, int linhas, int colunas, int flag)
{
int X_index[8]={-1,-1,0,1,1,1,0,-1};
int Y_index[8]={0,1,1,1,0,-1,-1,-1};
int neighbours[9]={0,0,0,0,0,0,0,0,0};
int i,j,total=0;
int ans=0;
int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
int fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
int index = fil * colunas + col;
if (fil>0 && col>0 && fil < linhas-1 && col < colunas-1)
{
d_changing1[index]=0;
for(i=0; i<8; i++)
{
neighbours[i]=GrayScale_[(fil+X_index[i])*colunas + (col+Y_index[i])];
total+=neighbours[i];
}

neighbours[8]=total;
for(j=0; j<7; j++)
{
if(neighbours[j]==0 && neighbours[j+1]==1)
ans=ans+1;
}

if(neighbours[7]==0 && neighbours[0]==1)
ans=ans+1;
if(flag!=1)
{
if(GrayScale_[fil*colunas + col]==1 && neighbours[8]>=2 && neighbours[8]<=6
&& ans==1 && neighbours[0]*neighbours[2]*neighbours[4]==0
&& neighbours[2]*neighbours[4]*neighbours[6]==0)
{
d_changing1[index]=1;
cont[flag]=1;
}
}
else
{
if(GrayScale_[fil*colunas + col]==1 && neighbours[8]>=2 && neighbours[8]<=6
&& ans==1 && neighbours[0]*neighbours[2]*neighbours[6]==0
&& neighbours[0]*neighbours[4]*neighbours[6]==0)
{
d_changing1[index]=1;
cont[flag]=1;
}
}
}
}