#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef TIME
#define COMM 1
#elif NOTIME
#define COMM 0
#endif

#define MASK_WIDTH 5
#define TILE_WIDTH 32
#define GPU 1
#define COMMENT "skeletization_GPU"
#define RGB_COMPONENT_COLOR 255


typedef struct {
unsigned char red, green, blue;
} PPMPixel;

typedef struct {
int x, y;
PPMPixel *data;
} PPMImage;

typedef struct {
int x, y;
} Par;

double time_total;
__global__ void Update(int *GrayScale_, int *d_changing1, int linhas, int colunas)
{
int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
int fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
int index = fil * colunas + col;

if (fil>0 && col>0 && fil < linhas-1 && col < colunas-1)
{
if(d_changing1[index]==1)
GrayScale_[index]=0;
}
}