#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mandelKernel(double planoFactorXd, double planoFactorYd, double planoVxd, double planoVyd, int maxIteracionesd, unsigned int *coloresd, int img_width, int img_height, int num_processes, int my_pid) {
int columna, fila;
double X, Y;
double pReal = 0.0;
double pImag = 0.0;
double pRealAnt, pImagAnt, distancia;
// Determinar pixel
columna = blockIdx.x * blockDim.x + threadIdx.x;
fila = blockIdx.y * blockDim.y + threadIdx.y;

int real_row = (fila * num_processes) + my_pid;

X = (planoFactorXd * (double)columna) + planoVxd;
Y = (planoFactorYd * ((double)(img_height - 1) - (double)real_row)) + planoVyd;
int i = 0;
do {
pRealAnt = pReal;
pImagAnt = pImag;
pReal = ((pRealAnt * pRealAnt) - (pImagAnt * pImagAnt)) + X;
pImag = (2.0 * (pRealAnt * pImagAnt)) + Y;
i++;
distancia = pReal*pReal + pImag*pImag;
}while ((i < maxIteracionesd) && (distancia <= 4.0));
if(i == maxIteracionesd) i = 0;
coloresd[(fila * img_width) + columna] = i;
}