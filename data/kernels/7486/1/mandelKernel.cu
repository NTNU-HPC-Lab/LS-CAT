#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mandelKernel(double planoFactorXd, double planoFactorYd, double planoVxd, double planoVyd, int maxIteracionesd, unsigned int *coloresd, int img_width, int img_height, int num_processes, int my_pid, int rw) {
int columna, fila;
double X, Y;
double pReal = 0.0;
double pImag = 0.0;
double pRealAnt, pImagAnt, distancia;

// Determine pixel
columna = blockIdx.x * blockDim.x + threadIdx.x;
fila = (rw * MAX_ROWS_PER_KERNEL) + (blockIdx.y * blockDim.y) + threadIdx.y;
int real_row = (fila * num_processes) + my_pid;

if(real_row >= img_height)
return;

// Real pixel coords
X = (planoFactorXd * (double)columna) + planoVxd;
Y = (planoFactorYd * ((double)(img_height - 1) - (double)real_row)) + planoVyd;
int i = 0;
do {
pRealAnt = pReal;
pImagAnt = pImag;
pReal = ((pRealAnt * pRealAnt) - (pImagAnt * pImagAnt)) + X;
pImag = (2.0 * (pRealAnt * pImagAnt)) + Y;
i++;
distancia = pReal*pReal + pImag*pImag;
}while ((i < maxIteracionesd) && (distancia <= 4.0));
if(i == maxIteracionesd) i = 0;
coloresd[(fila * img_width) + columna] = i;
}