#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_unavailable(bool *available, int n_rows, const int *idx, int n_selected) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < n_selected) {
available[idx[tid]] = false;
}
}