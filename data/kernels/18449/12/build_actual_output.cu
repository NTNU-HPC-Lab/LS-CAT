#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void build_actual_output(int *output, int n_rows, int k, const int *idx_labels, const int64_t *indices) {
int element = threadIdx.x + blockDim.x * blockIdx.x;
if (element >= n_rows * k) return;

int ind = (int)indices[element];
output[element] = idx_labels[ind];
}