#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void Pi_GPU(float *x, float *y, int *totalCounts, int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x; // номер элемента
int threadCount = gridDim.x * blockDim.x; //cмещение

int countPoints = 0;
for (int i = idx; i < N; i += threadCount) {
if (x[i] * x[i] + y[i] * y[i] < 1) {
countPoints++;
}
}
atomicAdd(totalCounts, countPoints); // каждый поток суммирует в переменную
}