#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Multiply_Matrix_GPU(float* A, float* B, float* C , int BLOCK_SIZE , int N) {
// Индекс блока
int bx = blockIdx.x;
int by = blockIdx.y;

// Индекс нити
int tx = threadIdx.x;
int ty = threadIdx.y;

float total = 0.0;
int ia = N * BLOCK_SIZE * by + N * ty;
int ib = BLOCK_SIZE * bx + tx;

for (int k = 0; k < N; k++) {
total += A[ia + k] * B[ib + k * N];
}
int ic = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;

//Результирующая матрица
C[ic + N * ty + tx] = total;
}