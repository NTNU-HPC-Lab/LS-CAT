#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
__shared__ float data[16*15];
__shared__ float hist[32*13];
__shared__ float gauss[16];
const int tx = threadIdx.x;
const int bx = blockIdx.x;
for (int i=0;i<13;i++)
hist[i*32+tx] = 0.0f;
__syncthreads();
float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
if (tx<15)
gauss[tx] = exp(i2sigma2*(tx-7)*(tx-7));
int xp = (int)(d_Sift[bx + 0*maxPts] - 6.5f);
int yp = (int)(d_Sift[bx + 1*maxPts] - 6.5f);
int px = xp & 15;
int x = tx - px;

for (int y=0;y<15;y++) {
int memPos = 16*y + x;
int xi = xp + x;
int yi = yp + y;
if (xi<0) xi = 0;
if (xi>=w) xi = w-1;
if (yi<0) yi = 0;
if (yi>=h) yi = h-1;
if (x>=0 && x<15)
data[memPos] = g_Data[yi*w+xi];
}
__syncthreads();
for (int y=1;y<14;y++) {
int memPos = 16*y + x;
if (x>=1 && x<14) {
float dy = data[memPos+16] - data[memPos-16];
float dx = data[memPos+1]  - data[memPos-1];
int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
if (bin==32)
bin = 0;
float grad = sqrtf(dx*dx + dy*dy);
hist[32*(x-1)+bin] += grad*gauss[x]*gauss[y];
}
}
__syncthreads();
for (int y=0;y<5;y++)
hist[y*32+tx] += hist[(y+8)*32+tx];
__syncthreads();
for (int y=0;y<4;y++)
hist[y*32+tx] += hist[(y+4)*32+tx];
__syncthreads();
for (int y=0;y<2;y++)
hist[y*32+tx] += hist[(y+2)*32+tx];
__syncthreads();
hist[tx] += hist[32+tx];
__syncthreads();
if (tx==0)
hist[32] = 6*hist[0] + 4*(hist[1]+hist[31]) + (hist[2]+hist[30]);
if (tx==1)
hist[33] = 6*hist[1] + 4*(hist[2]+hist[0]) + (hist[3]+hist[31]);
if (tx>=2 && tx<=29)
hist[tx+32] = 6*hist[tx] + 4*(hist[tx+1]+hist[tx-1]) +
(hist[tx+2]+hist[tx-2]);
if (tx==30)
hist[62] = 6*hist[30] + 4*(hist[31]+hist[29]) + (hist[0]+hist[28]);
if (tx==31)
hist[63] = 6*hist[31] + 4*(hist[0]+hist[30]) + (hist[1]+hist[29]);
__syncthreads();
float v = hist[32+tx];
hist[tx] = (v>hist[32+((tx+1)&31)] && v>=hist[32+((tx+31)&31)] ? v : 0.0f);
__syncthreads();
if (tx==0) {
float maxval1 = 0.0;
float maxval2 = 0.0;
int i1 = -1;
int i2 = -1;
for (int i=0;i<32;i++) {
float v = hist[i];
if (v>maxval1) {
maxval2 = maxval1;
maxval1 = v;
i2 = i1;
i1 = i;
} else if (v>maxval2) {
maxval2 = v;
i2 = i;
}
}
float val1 = hist[32+((i1+1)&31)];
float val2 = hist[32+((i1+31)&31)];
float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
d_Sift[bx + 5*maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);
if (maxval2<0.8f*maxval1)
i2 = -1;
if (i2>=0) {
float val1 = hist[32+((i2+1)&31)];
float val2 = hist[32+((i2+31)&31)];
float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
d_Sift[bx + 6*maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
} else
d_Sift[bx + 6*maxPts] = i2;
}
}