#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
int i = __mul24(blockIdx.x, POSBLK_SIZE) + threadIdx.x;
if (i>=numPts)
return;
int p = d_Ptrs[i];
//if (p<w+1 || p>=(w*h-w-1))
//  printf("ComputePositions: read error\n");
float val[7];
val[0] = g_Data2[p];
val[1] = g_Data2[p-1];
val[2] = g_Data2[p+1];
float dx = 0.5f*(val[2] - val[1]);
float dxx = 2.0f*val[0] - val[1] - val[2];
val[3] = g_Data2[p-w];
val[4] = g_Data2[p+w];
float dy = 0.5f*(val[4] - val[3]);
float dyy = 2.0f*val[0] - val[3] - val[4];
val[5] = g_Data3[p];
val[6] = g_Data1[p];
float ds = 0.5f*(val[6] - val[5]);
float dss = 2.0f*val[0] - val[5] - val[6];
float dxy = 0.25f*
(g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
float dxs = 0.25f*
(g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
float dys = 0.25f*
(g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);
float idxx = dyy*dss - dys*dys;
float idxy = dys*dxs - dxy*dss;
float idxs = dxy*dys - dyy*dxs;
float idyy = dxx*dss - dxs*dxs;
float idys = dxy*dxs - dxx*dys;
float idss = dxx*dyy - dxy*dxy;
float det = idxx*dxx + idxy*dxy + idxs*dxs;
float idet = 1.0f / det;
float pdx = idet*
(idxx*dx + idxy*dy + idxs*ds);
float pdy = idet*
(idxy*dx + idyy*dy + idys*ds);
float pds = idet*
(idxs*dx + idys*dy + idss*ds);
if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f){
pdx = __fdividef(dx, dxx);
pdy = __fdividef(dy, dyy);
pds = __fdividef(ds, dss);
}
float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
d_Sift[i+0*maxPts] = (p%w) + pdx;
d_Sift[i+1*maxPts] = (p/w) + pdy;
d_Sift[i+2*maxPts] = d_ConstantA[0] * exp2f(pds*d_ConstantB[0]);
d_Sift[i+3*maxPts] = val[0] + dval;
float tra = dxx + dyy;
det = dxx*dyy - dxy*dxy;
d_Sift[i+4*maxPts] = __fdividef(tra*tra, det);
}