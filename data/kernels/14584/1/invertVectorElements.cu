#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void invertVectorElements(float* vector, int n)
{
int i = threadIdx.x;
if (i < n)
{
vector[i] = 1.0f / vector[i];
}
}