#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inverse_transform(float *in, float *out, int height, int width) {
// block elements
int my_x, k, t;
my_x = blockIdx.x * blockDim.x + threadIdx.x;

// iterate through each element, going from frequency to time domain
for (k = 0; k < height; k++) {
// difference, which will be used to subtract off
float realSum = 0;
// iterate through the input element
for (t = 0; t < width; t++) {
float angle = 2 * M_PI * (my_x * height + t) * (my_x * height + k) / height;
realSum += in[my_x * height + t] * cos(angle);
}
out[my_x * height + k] = (realSum / height);
}
}