#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fourier_transform(float *in, float *out, int height, int width, int blockConfig) {
// block elements and function variables
int my_x, k, t;
my_x = blockIdx.x * blockDim.x + threadIdx.x;

// iterate through each element, going from frequency to time domain
for (k = 0; k < height; k++) {
// difference, which will be used to subtract off
float realSum = 0.0;
// iterate through the input element
for (t = 0; t < width; t++) {
// calculate the angle and update the sum
float angle = 2 * M_PI * (my_x * height + t) * (my_x * width + k) / height;
realSum += in[my_x * height + t] * cos(angle);
}
// each output element will be the current sum for that index
out[my_x * height + k] = realSum;
}
}