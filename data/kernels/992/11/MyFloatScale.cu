#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MyFloatScale(float *a, float scale, int size) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = threadID; i < size; i += numThreads)
{
a[i] *= scale;
}
}