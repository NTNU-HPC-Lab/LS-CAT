#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillZeros(float* buf, int size) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = threadID; i < size; i += numThreads)
{
buf[i] = 0.0f;
}
}