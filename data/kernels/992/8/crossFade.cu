#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void crossFade(float* out1, float* out2, int numFrames){
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
float fn = float(threadID) / (numFrames - 1.0f);
out1[threadID * 2] = out1[threadID * 2] * (1.0f - fn) + out2[threadID * 2] * fn;
out1[threadID * 2 + 1] = out1[threadID * 2 + 1] * (1.0f - fn) + out2[threadID * 2 + 1] * fn;
}