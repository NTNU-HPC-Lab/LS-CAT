#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void interleave(float* input, float* output, int size) {
const int numThreads = blockDim.x * gridDim.x;
const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = threadID; i < size; i += numThreads) {
output[2 * i] = input[i];
output[2 * i + 1] = input[size + 2 + i];
}
}