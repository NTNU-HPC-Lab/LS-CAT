#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void timeDomainConvolutionNaive(float* ibuf, float* rbuf, float* obuf, long long oframes, long long rframes, int ch, float gain) {
int threadID = blockIdx.x * blockDim.x + threadIdx.x;
float value = 0;
for (int k = 0; k < rframes; k++) {
value += ibuf[threadID - k] * rbuf[k];
}
obuf[threadID * 2 + ch] = value * gain;

}