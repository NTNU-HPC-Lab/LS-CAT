#include "hip/hip_runtime.h"
#include "includes.h"
/* Programmaufruf mit 2 Argumenten:
1. Größe des Gitters (mit Rand): Nx+2 (= Ny+2)
2. Dimension eines Cuda-Blocks: dim_block (findet nur Anwendung, wenn Nx+2 > dim_block)
*/

/*
Globale Variablen stehen in allen Funktionen zur Verfuegung.
Achtung: Das gilt *nicht* fuer Kernel-Funktionen!
*/
int Nx, Ny, npts;
int *active;

/*
Fuer die Koordinaten:
i = 0,1,...,Nx+1
j = 0,1,...,Ny+1
wird der fortlaufenden Index berechnet
*/
__global__ void vec_add_gpu(double *sum, double *w, double a, double *v, const int nx, const int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
if (ix>0 && ix<(nx+1) && iy>0 && iy<(ny+1)) // Bedingung, dass nur innere Punkte berechnet werden
{
unsigned int idx = iy*(blockDim.x * gridDim.x) + ix;
sum[idx] = w[idx] + a*v[idx];
}
}