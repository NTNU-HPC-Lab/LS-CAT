#include "hip/hip_runtime.h"
#include "includes.h"
/* Programmaufruf mit 2 Argumenten:
1. Größe des Gitters (mit Rand): Nx+2 (= Ny+2)
2. Dimension eines Cuda-Blocks: dim_block (findet nur Anwendung, wenn Nx+2 > dim_block)
*/

/*
Globale Variablen stehen in allen Funktionen zur Verfuegung.
Achtung: Das gilt *nicht* fuer Kernel-Funktionen!
*/
int Nx, Ny, npts;
int *active;

/*
Fuer die Koordinaten:
i = 0,1,...,Nx+1
j = 0,1,...,Ny+1
wird der fortlaufenden Index berechnet
*/
__global__ void laplace_2d_gpu(double *w, double *v, const int nx, const int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
if (ix>0 && ix<(nx+1) && iy>0 && iy<(ny+1)) // Bedingung, dass nur innere Punkte berechnet werden
{
unsigned int idx = iy*(blockDim.x * gridDim.x) + ix;
w[idx] = 4*v[idx] - (v[idx-1] + v[idx+1] + v[(idx-(gridDim.x*blockDim.x))] + v[(idx+(gridDim.x*blockDim.x))]);
}
}