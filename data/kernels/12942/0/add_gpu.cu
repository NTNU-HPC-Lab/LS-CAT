#include "hip/hip_runtime.h"
#include "includes.h"


#define N 10

__global__ void add_gpu( int *a, int *b, int *c ) {
int tid = blockIdx.x; // handle the data at this index
if (tid < N)
c[tid] = a[tid] + b[tid];
}