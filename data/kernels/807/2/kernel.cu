#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int power_modulo_fast(long a, long b, long m)
{
long i;
long result = 1;
long  x = a%m;

for (i=1; i<=b; i<<=1)
{
x %= m;
if ((b&i) != 0)
{
result *= x;
result %= m;
}
x *= x;
}

return result;
}
__device__ float generate( hiprandState* globalState, int ind )
{
//int ind = threadIdx.x;
hiprandState localState = globalState[ind];
float RANDOM = hiprand_uniform( &localState );
globalState[ind] = localState;
return RANDOM;
}
__global__ void kernel(int p, bool* prime, hiprandState* globalState)
{
int i = blockIdx.x;
int a = 0;
while(*prime && i < PRECISION)
{
a = (generate(globalState, i%BLOCKS_NUBMER) * (p-2))+1;
if(power_modulo_fast(a, p-1, p) == 1)
{
i += BLOCKS_NUBMER;
}
else
{
*prime = false;
}
}

}