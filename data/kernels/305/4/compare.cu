#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void compare(char *d_a, int *d_b, int *d_c, int size, int lenString, int threshold) {

int my_id = blockDim.x * blockIdx.x + threadIdx.x;

if (my_id == position)
d_c[my_id] = 2;


if ((my_id < size) && (d_c[my_id] == 0) && (my_id != position)) {
int x, diffs = 0;

for (x = 0; x < lenString; x++) {
diffs += (bool)(d_a[(lenString*position)+x]^d_a[(my_id*lenString)+x]);

if (diffs > threshold)
break;
}

if (diffs <= threshold) {
d_b[position] += d_b[my_id];
d_c[my_id] = 1;
}
}
}