#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void anyLeft(int *d_c, int *remaining, int size) {
int my_id = blockDim.x * blockIdx.x + threadIdx.x;
if((d_c[my_id] == 0) && (my_id < size)) {
*remaining = 0;
}
}