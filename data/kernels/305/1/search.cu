#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void search(int *d_b, int *d_c, int size) {
int my_id = blockDim.x * blockIdx.x + threadIdx.x;
if((d_c[my_id] == 0) && (d_b[my_id] == largest) && (my_id < size)) {
position = my_id;
}
}