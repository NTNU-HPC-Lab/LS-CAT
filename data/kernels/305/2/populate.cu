#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void populate (int *d_b, int *copy_db, int *d_c, int size, int *left) {
int n = 0;
*left = 1;	// reinitalized to false to check if all strings are merged

int my_id = blockDim.x * blockIdx.x + threadIdx.x;

if (my_id < size) {
n = abs((bool)d_c[my_id] - 1);
copy_db[my_id] = d_b[my_id] * n;
}
}