#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__device__ void cuda_select(int *db, int size) {
int my_id = blockDim.x * blockIdx.x + threadIdx.x;

if(my_id < size) {
if(db[2 * my_id] > db[2 * my_id + 1])
db[my_id] = db[2 * my_id];
else
db[my_id] = db[2 * my_id + 1];
}
}
__global__ void select(int *db, int size) {
int height = (int)ceil(log2((double)size));
int i = 0;

for(i = 0; i < height; i++) {
size = (int)ceil((double) size/2);
cuda_select(db, size);
}
largest = db[0];
}