#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline static float euclid_distance(int    numCoords, int    numObjs, int    numClusters, int    tid, int    clusterId, float *objects, float *clusters )
{
float ans=0.0;
for (int i = 0; i < numCoords; i++) {
ans += (objects[3*tid+i] - clusters[i + clusterId*3]) *
(objects[3*tid+i] - clusters[i + clusterId*3]);
}

return(ans);
}
__global__ static void find_nearest_cluster(int numCoords, int numObjs, int numClusters, float *objects, float *deviceClusters, int *membership, int *changedmembership )
{
extern __shared__ float sharedMem[];
float *sh_Clusters = sharedMem;
float *sh_Objects = (float*)&sh_Clusters[numClusters * 3];

for(int i = 0; i < numCoords * numClusters; i++) {
sh_Clusters[i] = deviceClusters[i];
}
__syncthreads();

unsigned int tid = threadIdx.x;
int objectId = blockDim.x * blockIdx.x + threadIdx.x;

while (objectId < numObjs) {
int   index, i;
float dist, min_dist;

for(int i = 0; i < numCoords; i++) {
sh_Objects[3*tid+i] = objects[3*objectId+i];
}

index = 0;
min_dist = euclid_distance(numCoords, numObjs, numClusters, tid,
0, sh_Objects, sh_Clusters);

for (i=1; i<numClusters; i++) {
dist = euclid_distance(numCoords, numObjs, numClusters, tid,
i, sh_Objects, sh_Clusters);
if (dist < min_dist) {
min_dist = dist;
index    = i;
}
}
if (membership[objectId] != index)
{
changedmembership[objectId] = 1;
membership[objectId] = index;

}
objectId += blockDim.x * gridDim.x;
}
}