#include "hip/hip_runtime.h"
#include "includes.h"
// cuDEBYE SOURCE CODE VERSION 1.5
// TO DO:
// - REWRITE TO DOUBLE PRECISION DISTANCE CALCULATIONS FOR BENCHMARKING
// - CONSIDER NOT CALLING SQRT (HISTOGRAM OF VALUE UNDER SQUARE -> problem with memory, no solution jet) IN KERNEL TO SAVE COMPUTATION TIME
// - USE INTEGER VALUES INSTEAD OF FLOAT AND CALCULATE IN FEMTO METERS INSTEAD OF ANGSTROM -> INTEGER OPERATIONS SHOULD REPLACE ROUND AND SINGLE PRECISION OPERATIONS WITH ACCEPTABLE ERROR
// - IMPLEMENT A CLEVER ALGORYTHM TO SET GRID AND BLOCK SIZE AUTOMATICALLY
// - BINARY FILE SUPPORT FOR FASTER INFORMATION EXCHANGE AND LESS MEMORY CONSUMPTION OR/AND PYTHON7MATLAB INTERFACE TO GET ARRAYS DIRECTLY
// - CREATE INTERFACE TO DISCUS (READ DISCUS STRUCTURES)
// - IMPLEMENT USAGE OF MORE GPU'S
// - MULTIPLE EMPTY LINES IN ASCII CAN CAUSE A CRASH DURING READING
// - HOST AND THRUST OPERATIONS ARE VERY INEFFICIENT (BUT FAST ENOUGH) -> MAYBE REWRITE THEM
// - ELIMINATE COMPILER WARNINGS FOR A MORE STABLE PROGRAM


////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// PREAMBLE: LIBARIES AND USEFULL BASIC FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Include cuda libaries for parallel computing

// Thrust libaries from the cuda toolkit for optimized vector operations

// Libaries for input and output streams for display results and read and write files.
// Better than the old printf shit
using namespace std;	// Normally all stream functions have to called via prefix std:: -> So functions can called withaout prefix (Example: std::cout -> cout)

// Libary for measuring calculation time

// define the mathematical constant pi
# define PI 3.14159265358979323846

// Function to check if input file parsed via commandline exists
__global__ void atomicScatter(int type1, int type2, int size_K, double *occ, double *beq, double *K, double *a, double *b, double *c, double *ffoobb) {
// Kernel is executed for each K/TwoTheta (one dimensional grid)
int Idx = blockIdx.x*blockDim.x + threadIdx.x;
// Only execute if K/TwoTheta exists and is no phantom value, caused be discrete grid and block size.
if (Idx < size_K) {
double rp16pi2 = -0.006332573977646; // = (-1) * 1/(16*pi²)
double negativeHalfSquaredS = K[Idx] * K[Idx] * rp16pi2; // = -sin²(theta)/lambda², s = 2*sin(theta)/lambda = 1/d
// Calculate occupancy and debye-waller part of the prefactor
ffoobb[Idx] = occ[type1] * occ[type2];
ffoobb[Idx] = ffoobb[Idx] * exp(negativeHalfSquaredS*(beq[type1] + beq[type2]));
// Calculate atomic scattering factords from 11 parameter approximation.
double f1 = c[type1];
double f2 = c[type2];
for (int i = 0; i < 5; i++) {
f1 += a[type1 * 5 + i] * exp(b[type1 * 5 + i] * negativeHalfSquaredS);
f2 += a[type2 * 5 + i] * exp(b[type2 * 5 + i] * negativeHalfSquaredS);
}
// Complement prefactor with calculated scattering factors
ffoobb[Idx] = ffoobb[Idx] * f1*f2;
}
}