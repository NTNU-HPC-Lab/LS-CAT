#include "hip/hip_runtime.h"
#include "includes.h"


#ifdef __INTELLISENSE__
void __syncthreads();
#endif


// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations
void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;





__global__ void imageBlur(float *input, float *output, int width, int height) {

int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

int numcols = WIDTH;

float blur;

if (row <= height && col <= width && row > 0 && col > 0)
{
// weights
int		x1,
x3, x4, x5,
x7;

// blur
// 0.0 0.2 0.0
// 0.2 0.2 0.2
// 0.0 0.2 0.0

x1 = input[(row + 1) * numcols + col];			// up
x3 = input[row * numcols + (col - 1)];			// left
x4 = input[row * numcols + col];				// center
x5 = input[row * numcols + (col + 1)];			// right
x7 = input[(row + -1) * numcols + col];			// down

blur = (x1 * 0.2) + (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2) + (x7 * 0.2);

output[row * numcols + col] = blur;
}
}