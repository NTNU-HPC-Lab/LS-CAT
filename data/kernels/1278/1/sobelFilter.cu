#include "hip/hip_runtime.h"
#include "includes.h"


#ifdef __INTELLISENSE__
void __syncthreads();
#endif


// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations
void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;





__global__ void sobelFilter(float *input, float *output, int width, int height) {

int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

int numcols = WIDTH;

float gradient_h;
float gradient_v;
float gradient;
float thresh = 30;

if (row <= height && col <= width && row > 0 && col > 0)
{
int x0, x1, x2,
x3,	    x5,
x6, x7, x8;

// horizontal
// -1  0  1
// -2  0  2
// -1  0  1

// vertical
// -1 -2 -1
//  0  0  0
//  1  2  1

x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
x1 = input[(row + 1) * numcols + col];			// up
x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
x3 = input[row * numcols + (col - 1)];			// left
x5 = input[row * numcols + (col + 1)];			// right
x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
x7 = input[(row + -1) * numcols + col];			// down
x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


gradient_h = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);
gradient_v = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

if (gradient >= thresh)
{
gradient = 255;
}
else {
gradient = 0;
}
output[row * numcols + col] = gradient;
}
}