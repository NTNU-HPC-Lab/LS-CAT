#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize_clause_output_predict(int *clause_output, int *all_exclude)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

// Initialize clause output
for (int j = index; j < CLAUSES; j += stride) {
clause_output[j] = 1;
all_exclude[j] = 1;
}
}