#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_with_all_exclude(int *clause_output, int *all_exclude)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

// Initialize clause output
for (int j = index; j < CLAUSES; j += stride) {
if (all_exclude[j] == 1) {
clause_output[j] = 0;
}
}
}