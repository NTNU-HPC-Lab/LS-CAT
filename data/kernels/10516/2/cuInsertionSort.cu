#include "hip/hip_runtime.h"
#include "includes.h"
/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
* The modifications are
*      removed texture memory usage
*      removed split query KNN computation
*      added feature extraction with bilinear interpolation
*
* Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
*/

// Includes

// Constants used by the program
#define BLOCK_DIM                      16


//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//

/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist        distance matrix
* @param ind         index matrix
* @param width       width of the distance matrix and of the index matrix
* @param height      height of the distance matrix and of the index matrix
* @param k           number of neighbors to consider
*/


/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
* @param k       number of neighbors to consider
*/


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//


/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void cuInsertionSort(float *dist, int *ind, int width, int height, int k){

// Variables
int l, i, j;
float *p_dist;
int   *p_ind;
float curr_dist, max_dist;
int   curr_row,  max_row;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){
// Pointer shift, initialization, and max value
p_dist   = dist + xIndex;
p_ind    = ind  + xIndex;
max_dist = p_dist[0];
p_ind[0] = 1;

// Part 1 : sort kth firt elementZ
for (l=1; l<k; l++){
curr_row  = l * width;
curr_dist = p_dist[curr_row];
if (curr_dist<max_dist){
i=l-1;
for (int a=0; a<l-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=l; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
} else {
p_ind[l*width] = l+1;
}
max_dist = p_dist[curr_row];
}

// Part 2 : insert element in the k-th first lines
max_row = (k-1)*width;
for (l=k; l<height; l++){
curr_dist = p_dist[l*width];
if (curr_dist<max_dist){
i=k-1;
for (int a=0; a<k-1; a++){
if (p_dist[a*width]>curr_dist){
i=a;
break;
}
}
for (j=k-1; j>i; j--){
p_dist[j*width] = p_dist[(j-1)*width];
p_ind[j*width]   = p_ind[(j-1)*width];
}
p_dist[i*width] = curr_dist;
p_ind[i*width]   = l+1;
max_dist             = p_dist[max_row];
}
}
}
}