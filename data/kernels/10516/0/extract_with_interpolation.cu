#include "hip/hip_runtime.h"
#include "includes.h"
/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
* The modifications are
*      removed texture memory usage
*      removed split query KNN computation
*      added feature extraction with bilinear interpolation
*
* Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
*/

// Includes

// Constants used by the program
#define BLOCK_DIM                      16


//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//

/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist        distance matrix
* @param ind         index matrix
* @param width       width of the distance matrix and of the index matrix
* @param height      height of the distance matrix and of the index matrix
* @param k           number of neighbors to consider
*/


/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
* @param k       number of neighbors to consider
*/


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//


/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void extract_with_interpolation( int nthreads, float *data, float *n_xy_coords, float *extracted_data, int n_max_coord, int channels, int height, int width) {

int x0, x1, y0, y1, nc;
float wx0, wx1, wy0, wy1;
int n, nd;
float x, y;

for (int index = blockIdx.x * blockDim.x + threadIdx.x;
index < (nthreads);
index += blockDim.x * gridDim.x) {
n = (index / n_max_coord);
nd = n * n_max_coord * channels;
x = n_xy_coords[index * 2];
y = n_xy_coords[index * 2 + 1];

x0 = static_cast<int>(floor(x));
x1 = x0 + 1;
y0 = static_cast<int>(floor(y));
y1 = y0 + 1;

x0 = x0 <= 0 ? 0 : (x0 >= (width - 1)  ? (width - 1) : x0);
y0 = y0 <= 0 ? 0 : (y0 >= (height - 1) ? (height - 1) : y0);
x1 = x1 <= 0 ? 0 : (x1 >= (width - 1)  ? (width - 1) : x1);
y1 = y1 <= 0 ? 0 : (y1 >= (height - 1) ? (height - 1) : y1);

wx0 = static_cast<float>(x1) - x;
wx1 = x - x0;
wy0 = static_cast<float>(y1) - y;
wy1 = y - y0;

if(x0 == x1){ wx0 = 1; wx1 = 0; }
if(y0 == y1){ wy0 = 1; wy1 = 0; }
for(int c=0; c < channels; c++) {
nc = (n * channels + c) * height;
// extracted_data[index * channels + c] = wy0 * wx0 * data[(nc + y0) * width + x0]
// extracted_data[nd + index % n_max_coord + n_max_coord * c] = index;
extracted_data[nd + index % n_max_coord + n_max_coord * c] = wy0 * wx0 * data[(nc + y0) * width + x0]
+ wy1 * wx0 * data[(nc + y1) * width + x0]
+ wy0 * wx1 * data[(nc + y0) * width + x1]
+ wy1 * wx1 * data[(nc + y1) * width + x1];
}
}
}