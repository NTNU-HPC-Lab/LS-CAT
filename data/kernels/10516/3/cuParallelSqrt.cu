#include "hip/hip_runtime.h"
#include "includes.h"
/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
* The modifications are
*      removed texture memory usage
*      removed split query KNN computation
*      added feature extraction with bilinear interpolation
*
* Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
*/

// Includes

// Constants used by the program
#define BLOCK_DIM                      16


//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//

/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist        distance matrix
* @param ind         index matrix
* @param width       width of the distance matrix and of the index matrix
* @param height      height of the distance matrix and of the index matrix
* @param k           number of neighbors to consider
*/


/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
* @param k       number of neighbors to consider
*/


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//


/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void cuParallelSqrt(float *dist, int width, int k){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
dist[yIndex*width + xIndex] = sqrt(dist[yIndex*width + xIndex]);
}