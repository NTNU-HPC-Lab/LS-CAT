#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel(float *output, float *input, int classSize, int height, int width)
{
int batchIndex = blockIdx.x;
int index = threadIdx.x;

while (index < height*width) {
int y = index / width;
int x = index % width;
if (y >= height)
break;

// calculate input starting index in cuda layout (B x H x W x C)
int inputStartIndex =
(height*width*classSize)*batchIndex +
(width*classSize)*y +
(classSize)*x;

float sum = 0;
for (int i = 0; i < classSize; i++) {
sum += __expf(input[inputStartIndex + i]);
}
sum = 1.0f / sum;

for (int i = 0; i < classSize; i++) {
// calculate output index in torch layout (B x C x H x W)
int outputIndex =
(classSize*height*width)*batchIndex +
(height*width)*i +
(width)*y +
x;
output[outputIndex] = logf(sum * __expf(input[inputStartIndex + i]));
}
index += blockDim.x;
}
}