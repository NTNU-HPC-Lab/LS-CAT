#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int classSize, int height, int width)
{
int batchIndex = blockIdx.x;
int index = threadIdx.x;

while (index < height*width) {
int y = index / width;
int x = index % width;
if (y >= height)
break;

// calculate output starting index in cuda layout (B x H x W x C)
int outputStartIndex =
(height*width*classSize)*batchIndex +
(width*classSize)*y +
(classSize)*x;

float sum = 0;
for (int i = 0; i < classSize; i++) {
sum += gradOutput[outputStartIndex + i];
}

for (int i = 0; i < classSize; i++) {
// calculate input index in torch layout (B x C x H x W)
int inputIndex =
(classSize*height*width)*batchIndex +
(height*width)*i +
(width)*y +
x;
gradInput[inputIndex] = gradOutput[outputStartIndex + i] - __expf(output[outputStartIndex + i]) * sum;
}
index += blockDim.x;
}
}