#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


// generate a random square matrix
__global__ void matMulKernel2(float* P, float* M, float* N, int width) {
__shared__ float Mds[2][2];
__shared__ float Nds[2][2];
int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by*2 + ty; int col = bx*2 + tx;
float pVal = 0;

for(int ph = 0; ph < width/2; ++ph) {
Mds[ty][tx] = M[row*width + ph*2 + tx];
Nds[ty][tx] = N[(ph*2 + ty)*width + col];
__syncthreads();
for(int k = 0; k < 2; ++k)
pVal += Mds[ty][k]*Nds[k][tx];
__syncthreads();
}
P[row*width + col] = pVal;
}