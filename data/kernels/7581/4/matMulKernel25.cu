#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


// generate a random square matrix
__global__ void matMulKernel25(float* P, float* M, float* N, int width) {
__shared__ float Mds25[25][25];
__shared__ float Nds25[25][25];
int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by*25 + ty; int col = bx*25 + tx;
float pVal = 0;

for(int ph = 0; ph < width/25; ++ph) {
Mds25[ty][tx] = M[row*width + ph*25 + tx];
Nds25[ty][tx] = N[(ph*25 + ty)*width + col];
__syncthreads();
for(int k = 0; k < 25; ++k)
pVal += Mds25[ty][k]*Nds25[k][tx];
__syncthreads();
}
P[row*width + col] = pVal;
}