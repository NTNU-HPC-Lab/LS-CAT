#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


// generate a random square matrix
__global__ void matMulKernel20(float* P, float* M, float* N, int width) {
__shared__ float Mds20[20][20];
__shared__ float Nds20[20][20];
int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by*20 + ty; int col = bx*20 + tx;
float pVal = 0;

for(int ph = 0; ph < width/20; ++ph) {
Mds20[ty][tx] = M[row*width + ph*20 + tx];
Nds20[ty][tx] = N[(ph*20 + ty)*width + col];
__syncthreads();
for(int k = 0; k < 20; ++k)
pVal += Mds20[ty][k]*Nds20[k][tx];
__syncthreads();
}
P[row*width + col] = pVal;
}