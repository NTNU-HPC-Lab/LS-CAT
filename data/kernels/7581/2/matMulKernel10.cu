#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


// generate a random square matrix
__global__ void matMulKernel10(float* P, float* M, float* N, int width) {
__shared__ float Mds10[10][10];
__shared__ float Nds10[10][10];
int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by*10 + ty; int col = bx*10 + tx;
float pVal = 0;

for(int ph = 0; ph < width/10; ++ph) {
Mds10[ty][tx] = M[row*width + ph*10 + tx];
Nds10[ty][tx] = N[(ph*10 + ty)*width + col];
__syncthreads();
for(int k = 0; k < 10; ++k)
pVal += Mds10[ty][k]*Nds10[k][tx];
__syncthreads();
}
P[row*width + col] = pVal;
}