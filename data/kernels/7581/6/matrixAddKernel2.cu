#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAddKernel2(float* ans, float* M, float* N, int size) {
int row = blockIdx.y*blockDim.y + threadIdx.y;
if(row < size) {
for(int i = 0; i < size; ++i)
ans[row*size + i] = M[row*size + i] + N[row*size + i];
}
}