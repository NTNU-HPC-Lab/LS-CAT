#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulKernel(float* ans, float* M, float* N, int size) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if(row < size && col < size) {
float pVal = 0;
for (int i = 0; i < size; ++i)
pVal += M[row*size + i] * N[i*size + col];
ans[row*size + col] = pVal;
}
}