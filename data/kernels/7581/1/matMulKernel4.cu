#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif


// generate a random square matrix
__global__ void matMulKernel4(float* P, float* M, float* N, int width) {
__shared__ float Mds4[4][4];
__shared__ float Nds4[4][4];
int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int row = by*4 + ty; int col = bx*4 + tx;
float pVal = 0;

for(int ph = 0; ph < width/4; ++ph) {
Mds4[ty][tx] = M[row*width + ph*4 + tx];
Nds4[ty][tx] = N[(ph*4 + ty)*width + col];
__syncthreads();
for(int k = 0; k < 4; ++k)
pVal += Mds4[ty][k]*Nds4[k][tx];
__syncthreads();
}
P[row*width + col] = pVal;
}