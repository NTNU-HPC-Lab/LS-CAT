#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAddKernel1(float* ans, float* M, float* N, int size) {
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;
if((row < size) && (col < size)) {
ans[row*size + col] = M[row*size + col] + N[row*size + col];
}
}