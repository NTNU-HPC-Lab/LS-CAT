#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAddKernel3(float* ans, float* M, float* N, int size) {
int col = blockIdx.x*blockDim.x + threadIdx.x;
if(col < size) {
for(int i = 0; i < size; ++i)
ans[i*size + col] = M[i*size + col] + N[i*size + col];
}
}