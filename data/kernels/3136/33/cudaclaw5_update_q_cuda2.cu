#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaclaw5_update_q_cuda2(int mbc, int mx, int my, int meqn, double dtdx, double dtdy, double* qold, double* fm, double* fp, double* gm, double* gp)
{
int ix = threadIdx.x + blockIdx.x*blockDim.x;
int iy = threadIdx.y + blockIdx.y*blockDim.y;

if (ix < mx && iy < my)
{
int x_stride = meqn;
int y_stride = (2*mbc + mx)*x_stride;
int I_q = (ix+mbc)*x_stride + (iy+mbc)*y_stride;
int mq;

for(mq = 0; mq < meqn; mq++)
{
int i = I_q+mq;
qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i])
- dtdy * (gm[i+y_stride] - gp[i]);
}
}
}