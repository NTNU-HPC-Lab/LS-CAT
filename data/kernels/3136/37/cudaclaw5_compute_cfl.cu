#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaclaw5_compute_cfl(int idir, int mx, int my, int meqn, int mwaves, int mbc, double dx, double dy, double dt, double *speeds, double* cflgrid)
{
#if 0
# from fortran_source/cudaclaw5_flux2.f */

c     # compute maximum wave speed for checking Courant number:
cfl1d = 0.d0
do 50 mw=1,mwaves
do 50 i=1,mx+1
c          # if s>0 use dtdx1d(i) to compute CFL,
c          # if s<0 use dtdx1d(i-1) to compute CFL:
cfl1d = dmax1(cfl1d, dtdx1d(i)*s(mw,i),
&                          -dtdx1d(i-1)*s(mw,i))
50       continue
#endif
/* Compute largest waves speeds, scaled by dt/dx,  on grid */


}