#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_dot(int N, double *a, double *b, double *c)
{
// __shared__ double localDot[threadsPerBlock];  /* Statically defined */
extern __shared__ double localDot[];
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int localIndex = threadIdx.x;

double localSum = 0;
while (ix < N)
{
localSum += a[ix] * b[ix];  /* Reduction is here */
ix += blockDim.x * gridDim.x;
}

/* Store sum computed by this thread */
localDot[localIndex] = localSum;

/* Wait for all threads to get to this point */
__syncthreads();

/* Every block should add up sum computed on
threads in the block */
int i = blockDim.x/2;
while (i != 0)
{
if (localIndex < i)
{
localDot[localIndex] += localDot[localIndex + i];
}
__syncthreads();
i /= 2;
}

/* Each block stores local dot product */
if (localIndex == 0)
c[blockIdx.x] = localDot[0];
}