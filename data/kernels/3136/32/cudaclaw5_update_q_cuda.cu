#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaclaw5_update_q_cuda(int mbc, double dtdx, double dtdy, double* qold, double* fm, double* fp, double* gm, double* gp)
{
int mq = threadIdx.z;
int x = threadIdx.x;
int x_stride = blockDim.z;
int y = threadIdx.y;
int y_stride = (blockDim.x + 2*mbc)*x_stride;
int i = mq + (x+mbc)*x_stride + (y+mbc)*y_stride;
qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i])
- dtdy * (gm[i+y_stride] - gp[i]);
}