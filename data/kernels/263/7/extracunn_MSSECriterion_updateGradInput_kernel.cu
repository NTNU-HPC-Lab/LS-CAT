#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extracunn_MSSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
int k = blockIdx.x;
float *gradInput_k = gradInput + k*dim;
float *input_k = input + k*dim;
float *target_k = target + k*dim;

__shared__ float buffer[MSSECRITERION_THREADS];

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;
float sum = 0.0;
// msse
buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float z = input_k[i] - target_k[i];
buffer[threadIdx.x] += z;
}
__syncthreads();


//reduce
if (threadIdx.x == 0)
{
sum = 0;
for (int i=0; i<blockDim.x; i++)
{
sum += buffer[i];
}
}

// gradInput
for (int i=i_start; i<i_end; i+=i_step)
gradInput_k[i] = norm*sum;
}