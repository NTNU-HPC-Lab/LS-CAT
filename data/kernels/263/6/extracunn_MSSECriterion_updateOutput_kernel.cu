#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extracunn_MSSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim)
{
__shared__ float buffer[MSSECRITERION_THREADS];
int k = blockIdx.x;
float *input_k = input + k*dim;
float *target_k = target + k*dim;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

// msse
buffer[threadIdx.x] = 0;
for (int i=i_start; i<i_end; i+=i_step)
{
float z = input_k[i] - target_k[i];
buffer[threadIdx.x] += z;
}
__syncthreads();


//reduce
if (threadIdx.x == 0)
{
*output = 0;
for (int i=0; i<blockDim.x; i++)
{
*output += buffer[i];
}
*output *= (*output);
*output /= (-2*dim*dim);
}
}