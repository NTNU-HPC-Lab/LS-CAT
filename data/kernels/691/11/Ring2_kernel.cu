#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void Ring2_kernel( float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w )
{
int id1 = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
if (id1 < size) {
// int y1 = id1 / w, x1 = id1 % w;
if (mask[id1] != 0) {

int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
for (int dx = -ring; dx <= ring; dx++)
for (int dy = -ring; dy <= ring; dy++)
{
int _x2 = x2 + dx, _y2 = y2 + dy;
if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h)
{
m[_y2 * w + _x2] = 1;
}
}
}
}

return ;
}