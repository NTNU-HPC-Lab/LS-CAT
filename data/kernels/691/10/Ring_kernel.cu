#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void Ring_kernel( float *A, float *BP, int *corrAB, float *M, int ring, int c, int h, int w )
{
int id1 = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
int ringSize  = 2*ring + 1;
int ringPatch = ringSize * ringSize;
if (id1 < size) {
int y1 = id1 / w, x1 = id1 % w;
int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
// int id2 = y2 * w + x2;

for (int dx = -ring; dx <= ring; dx++)
for (int dy = -ring; dy <= ring; dy++)
{
int pIdx = (dy + ring) * ringSize + (dx + ring);
int _x2 = x2 + dx, _y2 = y2 + dy;
if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h)
{
for (int dc = 0; dc < c; dc++) {
// M[(dc * size + y1 * w + x1) * ringPatch + pIdx] =
M[(dc * size + y1 * w) * ringPatch + pIdx * w + x1] =
BP[dc * size + _y2 * w + _x2];
}
}
}

}

return ;
}