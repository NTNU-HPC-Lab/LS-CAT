#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
c[i] = s[i]*a[i] + (1-s[i])*(b ? b[i] : 0);
}
}