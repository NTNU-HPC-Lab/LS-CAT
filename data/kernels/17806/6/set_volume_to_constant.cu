#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_volume_to_constant(float * out, float value)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
return;

long int id = (k * c_Size.y + j) * c_Size.x + i;

out[id] = value;
}