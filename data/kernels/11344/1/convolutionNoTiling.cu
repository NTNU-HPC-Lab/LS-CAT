#include "hip/hip_runtime.h"
#include "includes.h"





#define _USE_MATH_DEFINES

static void CheckCudaErrorAux(const char *, unsigned, const char *,
hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
__global__ void convolutionNoTiling(float *I, float *P, int channels, int width, int height) {

int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int depth = threadIdx.z;

if (col < width && row < height && depth < channels) {

// Evaluate convolution
float pValue = 0;

int startRow = row - maskRowsRadius;
int startCol = col - maskColumnsRadius;

for (int i = 0; i < maskRows; i++) {
for (int j = 0; j < maskColumns; j++) {
int currentRow = startRow + i;
int currentCol = startCol + j;

float iValue;

// Check for ghost elements
if (currentRow >= 0 && currentRow < height && currentCol >= 0 && currentCol < width) {
iValue = I[(currentRow * width + currentCol) * channels + depth];
}
else {
iValue = 0.0f;
}

pValue += iValue * deviceMaskData[i * maskRows + j];
}
}

// Salva il risultato dal registro alla global
P[(row * width + col) * channels + depth] = pValue;
}
}