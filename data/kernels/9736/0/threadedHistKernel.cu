#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void threadedHistKernel(int *threadedHist, int *arr, const int blockSize, const int valRange, const int threadBlockSize)
{
int val,
bid = blockIdx.x,
tid = threadIdx.x,
pid = bid*blockSize + tid;  //positional ID

// each thread takes info from its given info and increases the relevant position on the threadedHist
for (int i = 0; i < threadBlockSize; i++)
{
val = arr[pid*threadBlockSize + i];
threadedHist[valRange*pid + val]++;

}
}