#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void sumThreadedResultsKernel(long *dev_hist, int *dev_threadedHist, const int valRange, const int Blocks)
{
//e.g. tid from 0 to valRange-1, blocks = THREADS_PER_BLOCK * NO_BLOCKS
int tid = threadIdx.x;

for (int bl = 0; bl < Blocks; bl++)
{
dev_hist[tid] += dev_threadedHist[bl*valRange + tid];
}
}