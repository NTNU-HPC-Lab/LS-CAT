#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel0(int n, float a, float *x, float *y){

int i = blockIdx.x*blockDim.x + threadIdx.x;



//comment out this for-loop and uncomment the code in the main function for getting correct results
for (int i = 0; i < n; i++) {
x[i] = 1.0f;
y[i] = 2.0f;
}

if (i < n){
y[i] = a*x[i] + y[i];
}
}