#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel0(int n, float a, float *x, float *y){

int i = blockIdx.x*blockDim.x + threadIdx.x;


if (i < n){
y[i] = a*x[i] + y[i];
}
}