#include "hip/hip_runtime.h"
#include "includes.h"




using namespace std;

__global__ void matrixEuclideanDistanceKernelFast(float* in, float* out, int n, int m){
__shared__ float Ys[16][16];
__shared__ float Xs[16][16];

int bx = blockIdx.x, by = blockIdx.y;
int tx = threadIdx.x, ty = threadIdx.y;

int yBegin = by * 16 * m;
int xBegin = bx * 16 * m;

int yEnd = yBegin + m - 1, y, x, k, o;

float tmp, s = 0;

for (y = yBegin, x = xBegin;
y <= yEnd;
y += 16, x += 16){
Ys[ty][tx] = in[y + ty * m + tx];
Xs[tx][ty] = in[x + ty * m + tx];
__syncthreads();

for (k = 0; k<16; k++){
tmp = Ys[ty][k] - Xs[k][tx];
s += tmp * tmp;
}
__syncthreads();
}
o = by * 16 * n + ty * n + bx * 16 + tx;
out[o] = s;
}