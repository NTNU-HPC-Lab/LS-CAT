#include "hip/hip_runtime.h"
#include "includes.h"




using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = blockId * (blockDim.x * blockDim.y)
+ (threadIdx.y * blockDim.x)
+ threadIdx.x;
return threadId;
}
__global__ void matrixSquareElementWiseKernel(float* in, float* out, int n, int m){
extern __shared__ float Rs[];

int index = getGlobalIdx_2D_2D();
if (index < n*m){

out[index] = in[index] * in[index];

}
}