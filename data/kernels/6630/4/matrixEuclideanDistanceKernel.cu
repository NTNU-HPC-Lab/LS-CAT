#include "hip/hip_runtime.h"
#include "includes.h"




using namespace std;

__global__ void matrixEuclideanDistanceKernel(float* in, float* out, int n, int m){
extern __shared__ float Rs[];
float tmp, s;
int myRow = blockIdx.x*blockDim.x + threadIdx.x;
for (int r = 0; r<n; r++){ //outer loop
s = 0;
for (int i = 0; i <= m / 256; i++){
if (i * 256 + threadIdx.x < m)
Rs[i * 256 + threadIdx.x] = in[r*m + i * 256 + threadIdx.x];
}
__syncthreads();
for (int i = 0; i<m && myRow<n; i++){
tmp = Rs[i] - in[myRow*m + i];
s += tmp*tmp;
}
if (myRow<n)
out[myRow*n + r] = s;
__syncthreads();
}
}