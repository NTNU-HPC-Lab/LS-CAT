#include "hip/hip_runtime.h"
#include "includes.h"




using namespace std;

__global__ void matrixEuclideanDistanceKernelFastPartialOut(float* in_X, float* in_Y, float* out, int_least64_t n, int_least64_t m, int_least64_t start_out, int_least64_t end_out){
__shared__ float Ys[16][16];
__shared__ float Xs[16][16];

int_least64_t bx = blockIdx.x, by = blockIdx.y;
int_least64_t tx = threadIdx.x, ty = threadIdx.y;

int_least64_t yBegin = by * 16 * m;
int_least64_t xBegin = bx * 16 * m;

int_least64_t yEnd = yBegin + m - 1, y, x, k;
int_least64_t o;

float tmp, s = 0;

for (y = yBegin, x = xBegin;
y <= yEnd;
y += 16, x += 16){
Ys[ty][tx] = in_Y[y + ty * m + tx];
Xs[tx][ty] = in_X[x + ty * m + tx];
__syncthreads();

for (k = 0; k<16; k++){
tmp = Ys[ty][k] - Xs[k][tx];
s += tmp * tmp;
}
__syncthreads();
}

o = by * 16 * n + ty * n + bx * 16 + tx;
if (o >= start_out && o < end_out){
out[o - start_out] = s;
}
}