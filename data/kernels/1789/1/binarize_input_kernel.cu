#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (s >= size) return;
int i = 0;
float mean = 0;
for(i = 0; i < n; ++i){
mean += fabs(input[i*size + s]);
}
mean = mean / n;
for(i = 0; i < n; ++i){
binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
}
}