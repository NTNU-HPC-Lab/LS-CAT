#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= n) return;

int k = id % c;
id /= c;
int b = id;

int i;
int out_index = (k + c*b);
output[out_index] = 0;
for(i = 0; i < w*h; ++i){
int in_index = i + h*w*(k + b*c);
output[out_index] += input[in_index];
}
output[out_index] /= w*h;
}