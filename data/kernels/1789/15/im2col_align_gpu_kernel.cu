#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_align_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col, const int bit_align)
{
//__shared__ float tmp_s[1];


int index = blockIdx.x*blockDim.x + threadIdx.x;
for (; index < n; index += blockDim.x*gridDim.x) {
int w_out = index % width_col;
int h_index = index / width_col;
int h_out = h_index % height_col;
int channel_in = h_index / height_col;
int channel_out = channel_in * ksize * ksize;
int h_in = h_out * stride - pad;
int w_in = w_out * stride - pad;
//float* data_col_ptr = data_col;
//float* data_col_ptr_32 = data_col + (channel_out * bit_align + h_out * width_col + w_out) / 32;
//data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
//data_col_ptr += channel_out * bit_align + h_out * width_col + w_out;
float* data_col_ptr = &data_col[channel_out * bit_align + h_out * width_col + w_out];
const float* data_im_ptr = data_im;
data_im_ptr += (channel_in * height + h_in) * width + w_in;
for (int i = 0; i < ksize; ++i) {
for (int j = 0; j < ksize; ++j) {
int h = h_in + i;
int w = w_in + j;

float val = (h >= 0 && w >= 0 && h < height && w < width) ?
data_im_ptr[i * width + j] : 0;

int pre_out_index = index % (width_col*height_col);
int out_index = (channel_out + i*ksize + j) * bit_align + pre_out_index;// h_out * width_col + w_out;
data_col[out_index] = val;

//(*data_col_ptr) = val;
//dst_s[threadIdx.x] = val;
//tmp_s[0] = val;

//(*data_col_ptr) = (h >= 0 && w >= 0 && h < height && w < width) ?
//    data_im_ptr[i * width + j] : 0;

//float src_val = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;
//unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
//if (threadIdx.x % WARP_SIZE == 0) *((unsigned int*)data_col_ptr_32) = bit_mask;
// use atomicOr() // *dst_ptr |= (mask << (col_index % 8));
//data_col_ptr_32 += bit_align / 32;

//data_col_ptr += height_col * width_col;
data_col_ptr += bit_align;
}
}
}
}