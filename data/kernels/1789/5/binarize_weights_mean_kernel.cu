#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int f = i / size;
if (f >= n) return;
float mean = mean_arr_gpu[f];
binary[i] = (weights[i] > 0) ? mean : -mean;
}