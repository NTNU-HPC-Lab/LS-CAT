#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_one(float * prp_0,int sz)
{
// Thread index
int tx = threadIdx.x + blockIdx.x * blockDim.x;
int ty = threadIdx.y + blockIdx.y * blockDim.y;
int tz = threadIdx.z + blockIdx.z * blockDim.z;

prp_0[tz*sz*sz + ty*sz + tx] = 1.0f;
}