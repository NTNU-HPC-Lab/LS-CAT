#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_launch_cuda_native(float * scalar, float * vector, int sxy, int sx , int sy , int sz , int stride)
{
int id[3];

id[0] = threadIdx.x + blockIdx.x * blockDim.x;
id[1] = threadIdx.y + blockIdx.y * blockDim.y;
id[2] = threadIdx.z + blockIdx.z * blockDim.z;

if (id[0] >= sx) {return;}
if (id[1] >= sy) {return;}
if (id[2] >= sz) {return;}

float s = scalar[id[2]*sxy+id[1]*sx+id[0]];

float v[3];

v[0] = vector[id[2]*sxy+id[1]*sx+id[0] + 0*stride];
v[1] = vector[id[2]*sxy+id[1]*sx+id[0] + 1*stride];
v[2] = vector[id[2]*sxy+id[1]*sx+id[0] + 2*stride];

printf("Grid point from CUDA %d %d %d     scalar: %f  vector: %f %f %f \n",id[0],id[1],id[2],s,v[0],v[1],v[2]);
}