#include "hip/hip_runtime.h"
#include "includes.h"


//////////// Calculates weighting for assembling single element solution ///////////
// One weight is evaluated for each node
// Added back to global memory
__global__ void glob_sols( float *Le, float *w, float *u_glob, float *ue, int *cells, int num_cells)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;
int v;
float Lii, weight;

if(idx < num_cells && idy < blockDim.y){
v = cells[(idx*3) + idy];               // getting global vertex number
Lii = Le[(idx*9) + (idy*3) + idy];

weight = Lii/w[v];

atomicAdd(&u_glob[v], weight * ue[(idx*3) + idy]);
}
}