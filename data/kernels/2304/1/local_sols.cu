#include "hip/hip_runtime.h"
#include "includes.h"


//////////// Calculates weighting for assembling single element solution ///////////
// One weight is evaluated for each node
// Added back to global memory
__device__ void jacobi_iter( float *ue, float *up_glob, int *cells, float *temp1, int idx, int idy)
{
float ue_new;
int v;
int offset = 15*threadIdx.x;

/*
Le_shrd = &temp1[offset];
be_shrd = &temp1[offset + 9];
u_old  = &temp1[offset + 12];
*/

v = cells[(idx*3) + idy];

ue_new = temp1[(offset + 9) + idy];
temp1[(offset + 12) + idy] = up_glob[v];

__syncthreads();

ue_new -= temp1[offset + (idy*3) + ((idy+1)%3) ] * temp1[(offset + 12) + (idy+1) % 3];
ue_new -= temp1[offset + (idy*3) + ((idy+2)%3) ] * temp1[(offset + 12) +  (idy+2) % 3];

ue_new /= temp1[offset + (idy*3) + idy];

ue[(idx*3) + idy] = ue_new;
}
__device__ void elems_shared_cpy(float *Le, float *be, float *temp1, int idx, int idy){
int offset = 15*threadIdx.x;

// Le_shrd = &temp1[offset];
// be_shrd = &temp1[offset + 9];

temp1[(offset + 9) + idy] = be[(idx*3) + idy];
for(int i=0; i<3; i++){
temp1[offset + (idy*3) + i] = Le[(idx*9) + (idy*3) + i];
}
}
__global__ void local_sols( float *Le, float *be, float *ue, float *up_glob, int *cells, int num_cells)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
int idy = blockIdx.y*blockDim.y + threadIdx.y;
extern __shared__ float temp1[];

if(idx < num_cells && idy < blockDim.y){
elems_shared_cpy(Le, be, temp1, idx, idy);
__syncthreads();
jacobi_iter(ue, up_glob, cells, temp1, idx, idy);
}
}