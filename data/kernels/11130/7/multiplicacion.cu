#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplicacion( int *a, int *b, int *c, int n, int m, int l ) {
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

c[j+i*l] = 0;

for(int k=0 ; k < m ; k++ ){
c[j+i*l] += a[k+i*m] * b[j+k*l];
}
}