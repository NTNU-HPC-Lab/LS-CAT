#include "hip/hip_runtime.h"
#include "includes.h"
// hiprand
#define N 100
#define T 4


void llenarMatriz(int*);

__global__ void multiplicacion( int *a, int *b, int *c ) {
int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 2047
int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 2047

c[j+i*N] = 0; // 4,194,303

for(int k=0 ; k < N ; k++ ){
c[j+i*N] += a[k+i*N] * b[j+k*N];
}
}