#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aproximarPi( float *x, float *y, int *z, int tam) {
int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 2047
int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 2047
int index = j + i*tam; // 0 - 4194303

if( (x[index] * x[index] + y[index] * y[index]) <= 1.0f){
atomicAdd(z, 1);
}
}