#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcularBloques(int *matriz, int *u, int *resultado, int num_bloques, int nc, int m ){
int index1 = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 1
int index2 = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 1
int suma = 0;

for(int i=0 ; i < num_bloques ; i++){
suma = 0;
for(int l=0 ; l < nc ; l++){
suma += matriz[l+index1*nc] * u[index2+m*(l+i*nc)];
}
resultado[index2 + m*(index1+i*nc)] = suma;
}
}