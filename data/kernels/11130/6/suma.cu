#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void suma( int *a, int *b, int *c, int n, int m) {
int index = blockIdx.x + blockIdx.y * blockDim.y;
if(index < n*m){
c[index] = a[index] + b[index];
}
}