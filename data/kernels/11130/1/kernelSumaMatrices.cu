#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelSumaMatrices(float *a, float *b,int m, int n) {
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

while(i<m){
j = threadIdx.y + blockIdx.y*blockDim.y;
while(j<n){
a[i*n+j]+=b[i*n+j];
j+= blockDim.y*gridDim.y;
}
i+=blockDim.x*gridDim.x;
}
}