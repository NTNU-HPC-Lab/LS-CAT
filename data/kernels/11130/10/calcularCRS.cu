#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcularCRS(int *val, int *col_ind, int *row_ptr, int *u, int *resultado, int l ){
int i = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 9
int j = threadIdx.y + blockIdx.y*blockDim.y; // 0 - 9
int suma = 0;

for(int k = row_ptr[i]-1; k < row_ptr[i+1]-1; k++){
suma += val[k] * u[j + ( (col_ind[k]-1) * l) ];
}
resultado[j+i*l] = suma;
}