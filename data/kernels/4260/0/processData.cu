#include "hip/hip_runtime.h"
#include "includes.h"

unsigned char *pdata; // pointer to data content

__global__ void processData(unsigned char *Da, int* filter)
{
int tx = threadIdx.x;           // thread的x軸id
int bx = blockIdx.x;            // block的x軸id
int bn = blockDim.x;
int gid = bx * bn + tx;
__shared__ int sfilter[3][3];
__shared__ int sR[3][512];      // 每個block存上中下三行
__shared__ int sG[3][512];
__shared__ int sB[3][512];
__shared__ int sRsum[512];      // 每個block 最後512個sum
__shared__ int sGsum[512];
__shared__ int sBsum[512];

if (tx < 9)                     // 每個block 存filter 到 share memory
{
sfilter[tx / 3][tx % 3] = filter[tx];
}
__syncthreads();

if (bx == 0 || bx == 511 || tx == 0 || tx == 511)
{
// 邊界處理 --> 直接給原本值不動
sRsum[tx] = Da[gid * 3];
sGsum[tx] = Da[gid * 3 + 1];
sBsum[tx] = Da[gid * 3 + 2];
}

// 邊界處理(第1個block跟最後一個block不做)
if (bx != 0 && bx != 511)
{
// R, G, B個別將該Row(Block)運算會用到的上中下三行存入Share Memory
sR[0][tx] = Da[gid * 3 - 512 * 3];
sR[1][tx] = Da[gid * 3];
sR[2][tx] = Da[gid * 3 + 512 * 3];

sG[0][tx] = Da[gid * 3 - 512 * 3 + 1];
sG[1][tx] = Da[gid * 3 + 1];
sG[2][tx] = Da[gid * 3 + 512 * 3 + 1];

sB[0][tx] = Da[gid * 3 - 512 * 3 + 2];
sB[1][tx] = Da[gid * 3 + 2];
sB[2][tx] = Da[gid * 3 + 512 * 3 + 2];
__syncthreads();

// 邊界處理(每個block的的第一個值和最後一個值不做)
if (tx != 0 && tx != 511)
{
// R
sRsum[tx] = sR[0][tx - 1] * sfilter[0][0];
sRsum[tx] += sR[0][tx] * sfilter[0][1];
sRsum[tx] += sR[0][tx + 1] * sfilter[0][2];

sRsum[tx] += sR[1][tx - 1] * sfilter[1][0];
sRsum[tx] += sR[1][tx] * sfilter[1][1];
sRsum[tx] += sR[1][tx + 1] * sfilter[1][2];

sRsum[tx] += sR[2][tx - 1] * sfilter[2][0];
sRsum[tx] += sR[2][tx] * sfilter[2][1];
sRsum[tx] += sR[2][tx + 1] * sfilter[2][2];

// G
sGsum[tx] = sG[0][tx - 1] * sfilter[0][0];
sGsum[tx] += sG[0][tx] * sfilter[0][1];
sGsum[tx] += sG[0][tx + 1] * sfilter[0][2];

sGsum[tx] += sG[1][tx - 1] * sfilter[1][0];
sGsum[tx] += sG[1][tx] * sfilter[1][1];
sGsum[tx] += sG[1][tx + 1] * sfilter[1][2];

sGsum[tx] += sG[2][tx - 1] * sfilter[2][0];
sGsum[tx] += sG[2][tx] * sfilter[2][1];
sGsum[tx] += sG[2][tx + 1] * sfilter[2][2];

// B
sBsum[tx] = sB[0][tx - 1] * sfilter[0][0];
sBsum[tx] += sB[0][tx] * sfilter[0][1];
sBsum[tx] += sB[0][tx + 1] * sfilter[0][2];

sBsum[tx] += sB[1][tx - 1] * sfilter[1][0];
sBsum[tx] += sB[1][tx] * sfilter[1][1];
sBsum[tx] += sB[1][tx + 1] * sfilter[1][2];

sBsum[tx] += sB[2][tx - 1] * sfilter[2][0];
sBsum[tx] += sB[2][tx] * sfilter[2][1];
sBsum[tx] += sB[2][tx + 1] * sfilter[2][2];


sRsum[tx] /= filter[9];
sGsum[tx] /= filter[9];
sBsum[tx] /= filter[9];
// 大於255 或 小於0處理
if (sRsum[tx] > 255)
sRsum[tx] = 255;
else if (sRsum[tx] < 0)
sRsum[tx] = 0;

if (sGsum[tx] > 255)
sGsum[tx] = 255;
else if (sGsum[tx] < 0)
sGsum[tx] = 0;

if (sBsum[tx] > 255)
sBsum[tx] = 255;
else if (sBsum[tx] < 0)
sBsum[tx] = 0;
}
}

__syncthreads();

// 將R, G, B三個陣列值合併寫回一維陣列，以利輸出到檔案
Da[gid * 3] = sRsum[tx];
Da[gid * 3 + 1] = sGsum[tx];
Da[gid * 3 + 2] = sBsum[tx];
}