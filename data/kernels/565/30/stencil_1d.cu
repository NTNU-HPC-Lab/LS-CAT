#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int n, double *in, double *out)
{
/* allocate shared memory */
__shared__ double temp[THREADS_PER_BLOCK + 2*(RADIUS)];

/* calculate global index in the array */
int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
int localIndex = threadIdx.x + RADIUS;

/* return if my global index is larger than the array size */
if( globalIndex >= n ) return;

/* read input elements into shared memory */
temp[localIndex] = in[globalIndex];

/* code to handle the halos.  need to make sure we don't walk off the end
of the array */
if( threadIdx.x < RADIUS && globalIndex >= RADIUS )
{
temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
} /* end if */

if( threadIdx.x < RADIUS && globalIndex < (n - RADIUS) )
{
temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
} /* end if */

/* code to handle the boundary conditions */
if( globalIndex < RADIUS || globalIndex >= (n - RADIUS) )
{
out[globalIndex] = (double) globalIndex * ( (double)RADIUS*2 + 1) ;
return;
} /* end if */

double result = 0.0;

for( int i = -(RADIUS); i <= (RADIUS); i++ )
{
result += temp[localIndex + i];
} /* end for */

out[globalIndex] = result;
return;

}