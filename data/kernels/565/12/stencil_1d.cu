#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int n, double *in, double *out)
{
/* calculate global index in the array */
int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

/* return if my global index is larger than the array size */
if( globalIndex >= n ) return;

/* code to handle the boundary conditions */
if( globalIndex < RADIUS || globalIndex >= (n - RADIUS) )
{
out[globalIndex] = (double) globalIndex * ( (double)RADIUS*2 + 1) ;
return;
} /* end if */

double result = 0.0;

for( int i = globalIndex-(RADIUS); i <= globalIndex+(RADIUS); i++ )
{
result += in[i];
}

out[globalIndex] = result;
return;

}