#include "hip/hip_runtime.h"
#include "includes.h"
__global__	void	SampleConcentrationSlicedDev(float* concentration, uint slice,  const uint2*	cellStartEnd)
{
const	uint	cellid = gridDim.x*blockDim.x*slice + threadIdx.x*gridDim.x + blockIdx.x;

uint2	cellStEnd = cellStartEnd[cellid];

concentration[threadIdx.x*gridDim.x + blockIdx.x] = cellStEnd.y - cellStEnd.x;
}