#include "hip/hip_runtime.h"
#include "includes.h"
__global__	void	PreProcessVectorFieldDev(float3*	field, float width, float height, float minx, float miny)
{
uint	idx = threadIdx.x*gridDim.x + blockIdx.x;

float3	p1 = field[idx*2+0];
float3	p2 = field[idx*2+1];

p1.x -= minx;
p1.y -= miny;
p1.z = 0;

p1.x /= width;
p1.y /= height;

p2.x -= minx;
p2.y -= miny;
p2.z = 0;

p2.x /= width;
p2.y /= height;

field[idx*2+0] = p1;
field[idx*2+1] = p2;
}