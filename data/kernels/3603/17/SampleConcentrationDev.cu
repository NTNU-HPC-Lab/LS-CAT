#include "hip/hip_runtime.h"
#include "includes.h"
__global__	void	SampleConcentrationDev(float* concentration, const uint2*	cellStartEnd)
{
const	uint	cellid = gridDim.x*gridDim.y*threadIdx.x + blockIdx.y*gridDim.x + blockIdx.x;

uint2	cellStEnd = cellStartEnd[cellid];

concentration[cellid] = cellStEnd.y - cellStEnd.x;
}