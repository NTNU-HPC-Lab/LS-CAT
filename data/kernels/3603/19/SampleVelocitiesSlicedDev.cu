#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float length(float3 vec)
{
return	sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}
__device__ float length4(float4 vec)
{
return	sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}
__global__	void	SampleVelocitiesSlicedDev(float* velocities, const uint slice, const float4* vels_data, const uint2*	cellStartEnd,const uint* indices)
{
const	uint	cellid = gridDim.x*blockDim.x*slice + threadIdx.x*gridDim.x + blockIdx.x;

uint2	cellStEnd = cellStartEnd[cellid];

const uint	part_in_cell = cellStEnd.y - cellStEnd.x;

if(part_in_cell <= 0)
{
velocities[threadIdx.x*gridDim.x + blockIdx.x] = 0;
return;
}

float4	vel,p = make_float4(0,0,0,0);

for(uint	index = cellStEnd.x; index < cellStEnd.y; index++)
{
#ifndef	REORDER
uint	idx = indices[index];
vel = vels_data[idx];
#else
vel = vels_data[index];
#endif

p.x += vel.x;
p.y += vel.y;
p.z += vel.z;
}

velocities[threadIdx.x*gridDim.x + blockIdx.x] = length4(p) / part_in_cell;
}