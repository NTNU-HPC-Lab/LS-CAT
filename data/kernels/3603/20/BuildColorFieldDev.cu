#include "hip/hip_runtime.h"
#include "includes.h"
__global__	void	BuildColorFieldDev(float* data, uchar4* colors, float* minmax, uint xx, uint yy)
{
float	mn = minmax[0];
float	mx = minmax[1];

float	median = (mx - mn)/2.0f;

const uint idx = threadIdx.x*gridDim.x/yy/yy + blockIdx.x/xx;

float	val = data[idx];

uchar4	col;

#if	1

if(val < median)
{
float alpha = (val - mn)/(median - mn);

col.x = 0;
col.y = 255*(1-alpha);
col.z = 255*alpha;

}else
{
float alpha = (val - median)/(mx - median);

col.x = 255*alpha;
col.y = 0;
col.z = 255*(1-alpha);
}
#else

float	alpha = 1;

if(!(val < 0.1 || mn == mx || mx < 0.1))
alpha = val/(mx-mn);

col.x = 255*(1-alpha);
col.y = 255*(1-alpha);
col.z = 255*(1-alpha);

#endif
col.w = 255;

const	uint	col_idx = threadIdx.x*gridDim.x + blockIdx.x;

colors[col_idx] = col;

}