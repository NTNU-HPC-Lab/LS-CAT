#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiKernel(float *C, float *A, float *B, int Width) {

const int BLOCK_SIZE = 16; // NOTE: This must be similar to line 338
// block indexes
int bx = blockIdx.x;
int by = blockIdx.y;

// thread indexes
int tx = threadIdx.x;
int ty = threadIdx.y;

// int col = bx * TILE_WIDTH  + tx
// int row = by * TILE_WIDTH  + ty

// Dividing the matrices into sub sections
// Dividing the matrix A
int a_begin = Width * BLOCK_SIZE * by;
int a_end = a_begin + Width - 1;
int a_step = BLOCK_SIZE;

// Dividing the matrix B
int b_begin = BLOCK_SIZE * bx;
int b_step = BLOCK_SIZE * Width;

float temp_c = 0;

// loop throught the submatrices
for (int a = a_begin, b = b_begin; a <= a_end;
a += a_step, b += b_step) {
// sub matrices
__shared__ float sub_a[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float sub_b[BLOCK_SIZE][BLOCK_SIZE];

sub_a[ty][tx] = A[a + Width * ty + tx];
sub_b[ty][tx] = A[b + Width * ty + tx];

__syncthreads();


// loop unroll may not work on cuda if compilation level -O3
// effects cuda code as wll in the assignment
// sub matrix multiplication
#pragma unroll
for (int k = 0; k < BLOCK_SIZE; ++k) {
temp_c += sub_a[ty][k] * sub_b[k][tx];
}
// sync all the global threads running the computations
__syncthreads();
}
int c = Width * BLOCK_SIZE * by + BLOCK_SIZE * bx;
C[c + Width * ty + tx] = temp_c;
//    printf("kernel Done \n");
}