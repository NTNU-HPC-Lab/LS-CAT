#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 20
#define pi(x) printf("%d\n",x);
#define HANDLE_ERROR(err) ( HandleError( err, __FILE__, __LINE__ ) )
#define th_p_block  256


__global__ void dotPro(long n, float *vec1, float *vec2, float *vec3) {

__shared__ float cache[th_p_block];
unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int cacheIdx =  threadIdx.x;
float temp = 0;
while(tid < n)
{
temp += vec1[tid] * vec2[tid];
tid += blockDim.x * gridDim.x;
}

cache[cacheIdx] = temp;
__syncthreads();

// reduction
unsigned i = blockDim.x/2; // need the num threads to be a power of two (256 is okay)
while( i != 0 ){
if(cacheIdx < i){
cache[cacheIdx] += cache[cacheIdx + i ];
}

__syncthreads(); //sync threads in the current block
// power of two needed here
i = i/2;
}
if(cacheIdx == 0){
vec3[blockIdx.x] = cache[0];
}
//    if (tid < n) vec3[i] = vec1[i] * vec2[i];
}