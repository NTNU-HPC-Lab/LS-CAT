#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
int i;
float x;
int idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index across the blocks
for (i = idx; i < nbin; i += nthreads * nblocks) {
x = (i + 0.5) * step;
sum[idx] += 4.0 / (1.0 + x * x);
}
}