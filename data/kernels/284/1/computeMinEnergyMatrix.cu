#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMinEnergyMatrix(float *energy, float *min_energy, int height, int width) {
const int bid = blockIdx.x;
const int tid = threadIdx.x;
const int pos = bid * blockDim.x + tid;

extern __shared__ float shared_row_energy[];

if(pos < width) {
for(int i=tid; i<width; i += blockDim.x) {
shared_row_energy[i] = energy[i];
min_energy[i] = energy[i];
}
} else {
return;
}
__syncthreads();

float temp[4];
for(int i=1; i<height; i++) {
int k = 0;
for(int j=tid; j<width; j += blockDim.x) {
float l = (j==0) ? 999999999 :  shared_row_energy[j-1];
float m = shared_row_energy[j];
float r = (j==width-1) ? 999999999: shared_row_energy[j+1];

float minimum = energy[i*width + j] + min(l, min(m, r));
temp[k++] = minimum;
}
__syncthreads();
k = 0;
for(int j=tid; j<width; j += blockDim.x) {
shared_row_energy[j] = temp[k];
min_energy[i * width + j] = temp[k++];
}
}
}