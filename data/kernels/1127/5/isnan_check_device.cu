#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isnan_check_device(double *array, int size, bool *check) {
//
//  Description: Check for nan in array.

int idx = threadIdx.x + blockDim.x * blockIdx.x;

if (idx < size && ::isnan(array[idx])) {
*check = true;
}
}