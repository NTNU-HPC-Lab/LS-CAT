#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "CalcAngMom.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *AngMomx_d = NULL;
hipMalloc(&AngMomx_d, XSIZE*YSIZE);
double *AngMomy_d = NULL;
hipMalloc(&AngMomy_d, XSIZE*YSIZE);
double *AngMomz_d = NULL;
hipMalloc(&AngMomz_d, XSIZE*YSIZE);
double *GlobalAMx_d = NULL;
hipMalloc(&GlobalAMx_d, XSIZE*YSIZE);
double *GlobalAMy_d = NULL;
hipMalloc(&GlobalAMy_d, XSIZE*YSIZE);
double *GlobalAMz_d = NULL;
hipMalloc(&GlobalAMz_d, XSIZE*YSIZE);
double *Mh_d = NULL;
hipMalloc(&Mh_d, XSIZE*YSIZE);
double *Rho_d = NULL;
hipMalloc(&Rho_d, XSIZE*YSIZE);
double A = 2;
double Omega = 1;
double *Altitude_d = NULL;
hipMalloc(&Altitude_d, XSIZE*YSIZE);
double *Altitudeh_d = NULL;
hipMalloc(&Altitudeh_d, XSIZE*YSIZE);
double *lonlat_d = NULL;
hipMalloc(&lonlat_d, XSIZE*YSIZE);
double *areasT = NULL;
hipMalloc(&areasT, XSIZE*YSIZE);
double *func_r_d = NULL;
hipMalloc(&func_r_d, XSIZE*YSIZE);
int num = 1;
bool DeepModel = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
CalcAngMom<<<gridBlock,threadBlock>>>(AngMomx_d,AngMomy_d,AngMomz_d,GlobalAMx_d,GlobalAMy_d,GlobalAMz_d,Mh_d,Rho_d,A,Omega,Altitude_d,Altitudeh_d,lonlat_d,areasT,func_r_d,num,DeepModel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
CalcAngMom<<<gridBlock,threadBlock>>>(AngMomx_d,AngMomy_d,AngMomz_d,GlobalAMx_d,GlobalAMy_d,GlobalAMz_d,Mh_d,Rho_d,A,Omega,Altitude_d,Altitudeh_d,lonlat_d,areasT,func_r_d,num,DeepModel);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
CalcAngMom<<<gridBlock,threadBlock>>>(AngMomx_d,AngMomy_d,AngMomz_d,GlobalAMx_d,GlobalAMy_d,GlobalAMz_d,Mh_d,Rho_d,A,Omega,Altitude_d,Altitudeh_d,lonlat_d,areasT,func_r_d,num,DeepModel);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}