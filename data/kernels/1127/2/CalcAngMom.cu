#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcAngMom(double *AngMomx_d, double *AngMomy_d, double *AngMomz_d, double *GlobalAMx_d, double *GlobalAMy_d, double *GlobalAMz_d, double *Mh_d, double *Rho_d, double  A, double  Omega, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int     num, bool    DeepModel) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
double AMx, AMy, AMz;
double rx, ry, rz, r;

//calculate control volume
double zup, zlow, Vol;
zup  = Altitudeh_d[lev + 1] + A;
zlow = Altitudeh_d[lev] + A;
if (DeepModel) {
Vol = areasT[id] / pow(A, 2) * (pow(zup, 3) - pow(zlow, 3)) / 3;
}
else {
Vol = areasT[id] * (zup - zlow);
}

//radius vector
r  = (A + Altitude_d[lev]);
rx = r * func_r_d[id * 3 + 0];
ry = r * func_r_d[id * 3 + 1];
rz = r * func_r_d[id * 3 + 2];

//angular momentum r x p (total x and y over globe should ~ 0, z ~ const)
AMx = ry * Mh_d[id * 3 * nv + lev * 3 + 2] - rz * Mh_d[id * 3 * nv + lev * 3 + 1]
- Rho_d[id * nv + lev] * Omega * r * rz * cos(lonlat_d[id * 2 + 1])
* cos(lonlat_d[id * 2]);
AMy = -rx * Mh_d[id * 3 * nv + lev * 3 + 2] + rz * Mh_d[id * 3 * nv + lev * 3 + 0]
- Rho_d[id * nv + lev] * Omega * r * rz * cos(lonlat_d[id * 2 + 1])
* sin(lonlat_d[id * 2]);
AMz = rx * Mh_d[id * 3 * nv + lev * 3 + 1] - ry * Mh_d[id * 3 * nv + lev * 3 + 0]
+ Rho_d[id * nv + lev] * Omega * r * r * cos(lonlat_d[id * 2 + 1])
* cos(lonlat_d[id * 2 + 1]);
//AMx, AMy should go to zero when integrated over globe
// (but in practice, are just much smaller than AMz)

//total in control volume
AngMomx_d[id * nv + lev] = AMx * Vol;
AngMomy_d[id * nv + lev] = AMy * Vol;
AngMomz_d[id * nv + lev] = AMz * Vol;
}
}