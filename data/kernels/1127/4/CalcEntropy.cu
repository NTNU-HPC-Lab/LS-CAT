#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcEntropy(double *Entropy_d, double *pressure_d, double *temperature_d, double  Cp, double  Rd, double  A, double  P_Ref, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int     num, bool    DeepModel) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
double kappa = Rd / Cp;
double potT  = temperature_d[id * nv + lev] * pow(P_Ref / pressure_d[id * nv + lev], kappa);
double Sdens = Cp * log(potT);

//calculate control volume
double zup, zlow, Vol;
zup  = Altitudeh_d[lev + 1] + A;
zlow = Altitudeh_d[lev] + A;
if (DeepModel) {
Vol = areasT[id] / pow(A, 2) * (pow(zup, 3) - pow(zlow, 3)) / 3;
}
else {
Vol = areasT[id] * (zup - zlow);
}

//total energy in the control volume
Entropy_d[id * nv + lev] = Sdens * Vol;
}
}