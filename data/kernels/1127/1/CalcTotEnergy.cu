#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcTotEnergy(double *Etotal_d, double *GlobalE_d, double *Mh_d, double *W_d, double *Rho_d, double *temperature_d, double  Gravit, double  Cp, double  Rd, double  A, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int     num, bool    DeepModel) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
double Ek, Eint, Eg;
double wx, wy, wz;
double Cv = Cp - Rd;

//calculate control volume
double zup, zlow, Vol;
zup  = Altitudeh_d[lev + 1] + A;
zlow = Altitudeh_d[lev] + A;
if (DeepModel) {
Vol = areasT[id] / pow(A, 2) * (pow(zup, 3) - pow(zlow, 3)) / 3;
}
else {
Vol = areasT[id] * (zup - zlow);
}

//calc cartesian values of vertical wind
wx = W_d[id * nv + lev] * cos(lonlat_d[id * 2 + 1]) * cos(lonlat_d[id * 2]);
wy = W_d[id * nv + lev] * cos(lonlat_d[id * 2 + 1]) * sin(lonlat_d[id * 2]);
wz = W_d[id * nv + lev] * sin(lonlat_d[id * 2 + 1]);

//kinetic energy density 0.5*rho*v^2
Ek = 0.5
* ((Mh_d[id * 3 * nv + lev * 3 + 0] + wx) * (Mh_d[id * 3 * nv + lev * 3 + 0] + wx)
+ (Mh_d[id * 3 * nv + lev * 3 + 1] + wy) * (Mh_d[id * 3 * nv + lev * 3 + 1] + wy)
+ (Mh_d[id * 3 * nv + lev * 3 + 2] + wz) * (Mh_d[id * 3 * nv + lev * 3 + 2] + wz))
/ Rho_d[id * nv + lev];

//internal energy rho*Cv*T
Eint = Cv * temperature_d[id * nv + lev] * Rho_d[id * nv + lev];

//gravitation potential energy rho*g*altitude (assuming g = constant)
Eg = Rho_d[id * nv + lev] * Gravit * Altitude_d[lev];

//total energy in the control volume
Etotal_d[id * nv + lev] = (Ek + Eint + Eg) * Vol;

// printfn("E = %e\n",Etotal_d[id*nv+lev]);
}
}