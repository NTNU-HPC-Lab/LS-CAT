#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_mean(double* pressure_mean_d, double* pressure_d, double* Rho_mean_d, double* Rho_d, double* Mh_mean_d, double* Mh_d, double* Wh_mean_d, double* Wh_d, int     n_since_out, int     num) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
pressure_mean_d[id * nv + lev] =
1.0 / n_since_out
* (pressure_mean_d[id * nv + lev] * (n_since_out - 1) + pressure_d[id * nv + lev]);
Rho_mean_d[id * nv + lev] =
1.0 / n_since_out
* (Rho_mean_d[id * nv + lev] * (n_since_out - 1) + Rho_d[id * nv + lev]);
Mh_mean_d[3 * id * nv + 3 * lev + 0] =
1.0 / n_since_out
* (Mh_mean_d[3 * id * nv + 3 * lev + 0] * (n_since_out - 1)
+ Mh_d[3 * id * nv + 3 * lev] + 0);
Mh_mean_d[3 * id * nv + 3 * lev + 1] =
1.0 / n_since_out
* (Mh_mean_d[3 * id * nv + 3 * lev + 1] * (n_since_out - 1)
+ Mh_d[3 * id * nv + 3 * lev + 1]);
Mh_mean_d[3 * id * nv + 3 * lev + 2] =
1.0 / n_since_out
* (Mh_mean_d[3 * id * nv + 3 * lev + 2] * (n_since_out - 1)
+ Mh_d[3 * id * nv + 3 * lev + 2]);
Wh_mean_d[id * (nv + 1) + lev] =
1.0 / n_since_out
* (Wh_mean_d[id * (nv + 1) + lev] * (n_since_out - 1) + Wh_d[id * (nv + 1) + lev]);
if (lev == nv - 1) {
Wh_mean_d[id * (nv + 1) + lev + 1] =
1.0 / n_since_out
* (Wh_mean_d[id * (nv + 1) + lev + 1] * (n_since_out - 1)
+ Wh_d[id * (nv + 1) + lev + 1]);
}
}
}