#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcMass(double *Mass_d, double *GlobalMass_d, double *Rho_d, double  A, double *Altitudeh_d, double *lonlat_d, double *areasT, int     num, bool    DeepModel) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
//calculate control volume
double zup, zlow, Vol;
zup  = Altitudeh_d[lev + 1] + A;
zlow = Altitudeh_d[lev] + A;
if (DeepModel) {
Vol = areasT[id] / pow(A, 2) * (pow(zup, 3) - pow(zlow, 3)) / 3;
}
else {
Vol = areasT[id] * (zup - zlow);
}

//mass in control volume = density*volume
Mass_d[id * nv + lev] = Rho_d[id * nv + lev] * Vol;
}
}