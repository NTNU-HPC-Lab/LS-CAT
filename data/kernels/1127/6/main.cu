#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "rayleighHS.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *Mh_d = NULL;
hipMalloc(&Mh_d, XSIZE*YSIZE);
double *pressure_d = NULL;
hipMalloc(&pressure_d, XSIZE*YSIZE);
double *Rho_d = NULL;
hipMalloc(&Rho_d, XSIZE*YSIZE);
double *Altitude_d = NULL;
hipMalloc(&Altitude_d, XSIZE*YSIZE);
double surf_drag = 1;
double bl_sigma = 1;
double Gravit = 1;
double time_step = 1;
int num = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
rayleighHS<<<gridBlock,threadBlock>>>(Mh_d,pressure_d,Rho_d,Altitude_d,surf_drag,bl_sigma,Gravit,time_step,num);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
rayleighHS<<<gridBlock,threadBlock>>>(Mh_d,pressure_d,Rho_d,Altitude_d,surf_drag,bl_sigma,Gravit,time_step,num);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
rayleighHS<<<gridBlock,threadBlock>>>(Mh_d,pressure_d,Rho_d,Altitude_d,surf_drag,bl_sigma,Gravit,time_step,num);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}