#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rayleighHS(double *Mh_d, double *pressure_d, double *Rho_d, double *Altitude_d, double  surf_drag, double  bl_sigma, double  Gravit, double  time_step, int     num) {

int id  = blockIdx.x * blockDim.x + threadIdx.x;
int nv  = gridDim.y;
int lev = blockIdx.y;

if (id < num) {
double sigma;
double sigmab = bl_sigma;
double kf     = surf_drag;
double kv_hs;
double ps, pre;
double psm1;

//      Calculates surface pressure
psm1 = pressure_d[id * nv + 1]
- Rho_d[id * nv + 0] * Gravit * (-Altitude_d[0] - Altitude_d[1]);
ps = 0.5 * (pressure_d[id * nv + 0] + psm1);

pre   = pressure_d[id * nv + lev];
sigma = (pre / ps);

//      Momentum dissipation constant.
kv_hs = kf * max(0.0, (sigma - sigmab) / (1.0 - sigmab));

//      Update momenta
for (int k = 0; k < 3; k++)
Mh_d[id * 3 * nv + lev * 3 + k] =
Mh_d[id * 3 * nv + lev * 3 + k] / (1.0 + kv_hs * time_step);

// Wh_d[id * (nv + 1) + lev + k] = Wh_d[id * (nv + 1) + lev + k] / (1.0 + kv_hs * time_step);
}
}