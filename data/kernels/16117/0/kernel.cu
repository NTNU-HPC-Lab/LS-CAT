#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/



#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)




__global__ void kernel( int *a, int *b, int *c ) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < N) {
int idx1 = (idx + 1) % 256;
int idx2 = (idx + 2) % 256;
float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
c[idx] = (as + bs) / 2;
}
}