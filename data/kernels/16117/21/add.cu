#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < N) {
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}
}