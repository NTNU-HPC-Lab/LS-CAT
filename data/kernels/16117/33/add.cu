#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int *b, int *c)
{
int tid = blockIdx.x; //task ID? this is CPU code so we only got 1, CPU 0
if (tid < N)
c[tid] = a[tid] + b[tid];
}