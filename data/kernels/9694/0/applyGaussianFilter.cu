#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GAUSS_WIDTH 5
#define SOBEL_WIDTH 3

typedef struct images {
char *pType;
int width;
int height;
int maxValColor;
unsigned char *data;
} image;

/**
Reads the input file formatted as pnm. The actual implementation
supports only P5 type pnm images (grayscale).
*/
__global__ void applyGaussianFilter(unsigned char *input, unsigned char *output, float *kernel, int iHeight, int iWidth, int kWidth) {

int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

double sum = 0.0;

int halvedKW = kWidth / 2;

for (int i = -halvedKW; i <= halvedKW; i++) {
for (int j = -halvedKW; j <= halvedKW; j++) {
if ((x + j) < iWidth && (x + j) >= 0 && (y + i) < iHeight && (y + i) >= 0) {
int kPosX = (j + halvedKW);
int kPosY = (i + halvedKW);
sum = sum + (float)(input[(y + i) * iWidth + (x + j)]) * kernel[kPosY * kWidth + kPosX];
}
}
}

if (sum > 255.0)
sum = 255.0;

output[y * iWidth + x] = (unsigned char)sum;
}