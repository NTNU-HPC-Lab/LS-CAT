#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GAUSS_WIDTH 5
#define SOBEL_WIDTH 3

typedef struct images {
char *pType;
int width;
int height;
int maxValColor;
unsigned char *data;
} image;

/**
Reads the input file formatted as pnm. The actual implementation
supports only P5 type pnm images (grayscale).
*/
__global__ void applySobelFilter(unsigned char *in, unsigned char *intensity, float *direction, int ih, int iw) {

int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

int gx, gy;

if (x > 0 && x + 1 < iw && y > 0 && y + 1 < ih) {
gx =
1 * in[(y - 1) * iw + (x - 1)] + (-1) * in[(y - 1) * iw + (x + 1)] +
2 * in[y * iw + (x - 1)]	   + (-2) * in[y * iw + (x + 1)] +
1 * in[(y + 1) * iw + (x - 1)] + (-1) * in[(y + 1) * iw + (x + 1)];

gy =
1 * in[(y - 1) * iw + (x - 1)] +    2 * in[(y - 1) * iw + x] +    1 * in[(y - 1) * iw + (x + 1)] +
(-1) * in[(y + 1) * iw + (x - 1)] + (-2) * in[(y + 1) * iw + x] + (-1) * in[(y + 1) * iw + (x + 1)];

intensity[y * iw + x] = (unsigned char)sqrt((float)(gx) * (float)(gx) + (float)(gy) * (float)(gy));
direction[y * iw + x] = atan2((float)gy, (float)gx);
}
}