#include "hip/hip_runtime.h"
#include "includes.h"


typedef unsigned int  uint;
typedef unsigned char uchar;
typedef unsigned char VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;
texture<VolumeType, 3, hipReadModeNormalizedFloat> tex_block;
texture<float4,  1, hipReadModeElementType> texture_float_1D;

struct Ray
{
float3 o;   // origin
float3 d;   // direction
};

__device__
__device__ unsigned char myMAX(unsigned char a, unsigned char b)
{
if(a >= b)
return a;
else
return b;
}
__global__ void block_volume(unsigned char* image_p, unsigned char* dest_p, int srcWidth, int srcHeight, int srcDepth, int desWidth, int desHeight, int desDepth){


unsigned int tx = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int ty = blockIdx.y*blockDim.y + threadIdx.y;

if (tx >= desWidth || ty >= desHeight) return;

for(int i=0; i<desDepth; i++){
dest_p[i*desWidth*desHeight + ty*desHeight + tx] = 0;
unsigned char tempmax=0;

for(int z=i*4; z<=i*4+4; z++)
for(int y=ty*4; y<=ty*4+4; y++)
for(int x=tx*4; x<=tx*4+4; x++){
if(z>=srcDepth || y>=srcHeight || x>=srcWidth )
continue;
tempmax = myMAX(tempmax, image_p[z*srcWidth*srcHeight + y*srcHeight + x]);
}
dest_p[i*desWidth*desHeight + ty*desHeight + tx] = tempmax;
}

}