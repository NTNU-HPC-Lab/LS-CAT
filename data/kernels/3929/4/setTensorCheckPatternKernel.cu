#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setTensorCheckPatternKernel(unsigned int* data, unsigned int ndata) {
for (unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;i < ndata;i += blockDim.x*gridDim.x) {
data[i] = i;
}
}