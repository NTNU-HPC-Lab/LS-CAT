#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_decrypted(int *pDataPointer , int *pRandomData , int *pEncryptedData , long long int pSize)
{
long long int index = blockIdx.x * blockDim.x + threadIdx.x;
if( index <=(pSize /sizeof(int) ))
{
(*(pEncryptedData+index)) = (*(pDataPointer+ index))^(*(pRandomData+index));
}
else
return;
}