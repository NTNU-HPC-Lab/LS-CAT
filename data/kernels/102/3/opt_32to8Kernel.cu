#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length){
int idx = blockDim.x * blockIdx.x + threadIdx.x;

output[idx] = (uint8_t)((input[idx] < UINT8_MAX) * input[idx]) + (input[idx] >= UINT8_MAX) * UINT8_MAX;

__syncthreads();
}