#include "hip/hip_runtime.h"
#include "includes.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

__device__ int getPosition(int x, int y, int width, int margin, int pixelPosition)
{
return (x + (y * width)) * margin + pixelPosition;
}
__global__ void addEffect( unsigned char* output_img, unsigned char* input_img, int width, int height, int nbBlocks)
{
int lengthY = (int)(height/nbBlocks)+1;
int startY = blockIdx.x * lengthY;
int endY = blockIdx.x * lengthY + lengthY;

if( endY > height )
endY = height;

int lengthX = (int)(width/blockDim.x)+1;
int startX = threadIdx.x * lengthX;
int endX = threadIdx.x * lengthX + lengthX;

if( endX > width )
endX = width;

for( int x = startX; x < endX; x++ )
{
for( int y = startY; y < endY; y++ )
{
int currentIndex = getPosition(x, y, width, 3, 0);
if( (input_img[currentIndex] + input_img[currentIndex+1] + input_img[currentIndex+2])/3 < 20)
{
output_img[currentIndex] = input_img[currentIndex];
output_img[currentIndex+1] = input_img[currentIndex+1];
output_img[currentIndex+2] = input_img[currentIndex+2];

for( int i = -4; i <= 4; i++ )
{
for( int j = -4; j <= 4; j++ )
{
if( x+i < 0 || x+i > width || y+j < 0 || y+j > height )
continue;

int neighbourIndex = getPosition( x+i, y+j, width, 3, 0);

if( neighbourIndex < 0 || neighbourIndex + 2 > width*height*3)
continue;

output_img[neighbourIndex] = 0;
output_img[neighbourIndex+1] = 0;
output_img[neighbourIndex+2] = 0;
}

}
}
}

}

}