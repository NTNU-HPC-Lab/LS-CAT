#include "hip/hip_runtime.h"
#include "includes.h"
__device__ uint32_t RGBAPACK_8bit(float red, float green, float blue, uint32_t alpha)
{
uint32_t ARGBpixel = 0;

// Clamp final 10 bit results
red   = min(max(red,   0.0f), 255.0f);
green = min(max(green, 0.0f), 255.0f);
blue  = min(max(blue,  0.0f), 255.0f);

// Convert to 8 bit unsigned integers per color component
ARGBpixel = ((((uint32_t)red)   << 24) |
(((uint32_t)green) << 16) |
(((uint32_t)blue)  <<  8) | (uint32_t)alpha);

return  ARGBpixel;
}
__global__ void Passthru(uint32_t *srcImage,   size_t nSourcePitch, uint32_t *dstImage,   size_t nDestPitch, uint32_t width,       uint32_t height)
{
int x, y;
uint32_t yuv101010Pel[2];
uint32_t processingPitch = ((width) + 63) & ~63;
uint32_t dstImagePitch   = nDestPitch >> 2;
uint8_t *srcImageU8     = (uint8_t *)srcImage;

processingPitch = nSourcePitch;

// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
y = blockIdx.y *  blockDim.y       +  threadIdx.y;

if (x >= width)
return; //x = width - 1;

if (y >= height)
return; // y = height - 1;

// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
// if we move to texture we could read 4 luminance values
yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]);
yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]);

// this steps performs the color conversion
float luma[2];

luma[0]   = (yuv101010Pel[0]        & 0x00FF);
luma[1]   = (yuv101010Pel[1]        & 0x00FF);

// Clamp the results to RGBA
dstImage[y * dstImagePitch + x     ] = RGBAPACK_8bit(luma[0], luma[0], luma[0], constAlpha);
dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_8bit(luma[1], luma[1], luma[1], constAlpha);
}