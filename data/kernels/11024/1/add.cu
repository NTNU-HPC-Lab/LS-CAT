#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
int tid = threadIdx.x;
if (tid < N)
c[tid] = a[tid] + b[tid];
}