#include "hip/hip_runtime.h"
#include "includes.h"


#define N 10


__global__ void add(int *a, int *b, int *c)
{
int tid = blockIdx.x;
if (tid < N)
c[tid] = a[tid] + b[tid];
}