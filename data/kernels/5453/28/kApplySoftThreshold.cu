#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplySoftThreshold(float* mat, float alpha, float* target, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
float f = mat[i];
target[i] = f > 0 ? max(0., f - alpha) : min(0., f + alpha);
}
}