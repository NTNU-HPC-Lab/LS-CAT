#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCorrelate(float* source, float* kernel, float* dest, int width, int height, int kwidth, int kheight) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < width * height; i += numThreads) {
float sum = 0;
for (int w = -kwidth/2; w <= kwidth/2; w++) {
for (int h = -kheight/2; h <= (kheight)/2; h++) {
const int x = (i / height) + w;
const int y = (i % height) + h;
const int j = i + (w * height) + h;

if (x >= 0 && x < width && y >= 0 && y < height)
sum += source[j] * kernel[(kwidth * kheight / 2) + w * kheight + h];
}
}
dest[i] = sum;
}
}