#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < len; i += numThreads) {
dest[i] = alpha * mat[i];
}
}