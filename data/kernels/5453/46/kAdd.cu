#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < numEls; i += numThreads) {
dest[i] = a[i] + b[i];
}
}