#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorReduce(const float *global_input_data, float *global_output_data, const int numElements)
{
__shared__ float sdata[10];
__shared__  int sindice[10];

int tid = threadIdx.x;
int i = blockIdx.x * (blockDim.x ) + threadIdx.x;
sdata[tid] = global_input_data[i];
sindice[tid] = tid;
__syncthreads();

for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {

if (tid < s ) {
if (sdata[tid] > sdata[tid + s]) {
sdata[tid] = sdata[tid + s];
sindice[tid] = sindice[tid + s];
}
__syncthreads();
}
}

__syncthreads();

if (tid == 0) {
global_output_data[0] = sdata[0];

}

if (tid == 1) {
global_output_data[1] = sindice[0];

}

}