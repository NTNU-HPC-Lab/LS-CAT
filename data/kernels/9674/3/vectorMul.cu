#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorMul(const float *A, const float *B, float *C, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;


if (i < numElements)
{
C[i] = A[i] * B[i];
}
}