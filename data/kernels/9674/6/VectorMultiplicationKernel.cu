#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorMultiplicationKernel(int *array, int arrayCount)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < arrayCount)
{
array[idx] *= array[idx];
}
}