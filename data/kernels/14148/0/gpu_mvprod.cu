#include "hip/hip_runtime.h"
#include "includes.h"
/* Copyright (C) 2012  Ward Poelmans

This file is part of Hubbard-GPU.

Hubbard-GPU is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

Hubbard-GPU is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Hubbard-GPU.  If not, see <http://www.gnu.org/licenses/>.
*/


// number of threads in a block (must be multiple of 32)
#define NUMTHREADS 128

// the maximum size of the grid
#define GRIDSIZE 65535

// Helper macro to check CUDA return values
__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up, int rows_shared)
{
int index = threadIdx.x + blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x;

if(index < dim)
{
double result = Umat[index] * x[index];

int sv = index / NumDown; //__fdividef(index,NumDown);
int id = index % NumDown; // index - sv*NumDown;

extern __shared__ double shared[];

unsigned int *shared_ind = (unsigned int *) &shared[size_Up * rows_shared];

int s_sv = (blockDim.x * blockIdx.x + blockIdx.y * blockDim.x * gridDim.x)/NumDown;

if(threadIdx.x < rows_shared && (s_sv + threadIdx.x) < NumUp)
for(int i=0;i<size_Up;i++)
{
shared[i*rows_shared+threadIdx.x] = Up_data[s_sv + threadIdx.x + i*NumUp];

shared_ind[i*rows_shared+threadIdx.x] = Up_ind[s_sv + threadIdx.x + i*NumUp];
}

__syncthreads();

for(int i=0;i<size_Up;i++)
// result += Up_data[sv+i*NumUp] * x[id + NumDown*Up_ind[sv+i*NumUp]];
result += shared[sv-s_sv+i*rows_shared] * x[id + NumDown*shared_ind[sv-s_sv+i*rows_shared]];

for(int i=0;i<size_Down;i++)
result += Down_data[id+i*NumDown] * x[sv*NumDown + Down_ind[id+i*NumDown]];

y[index] = alpha * y[index] + result;
}
}