#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "gpu_mvprod.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
double *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
double alpha = 2;
int NumUp = 1;
int NumDown = 1;
int dim = 2;
double *Umat = NULL;
hipMalloc(&Umat, XSIZE*YSIZE);
double *Down_data = NULL;
hipMalloc(&Down_data, XSIZE*YSIZE);
unsigned int *Down_ind = NULL;
hipMalloc(&Down_ind, XSIZE*YSIZE);
int size_Down = XSIZE*YSIZE;
double *Up_data = NULL;
hipMalloc(&Up_data, XSIZE*YSIZE);
unsigned int *Up_ind = NULL;
hipMalloc(&Up_ind, XSIZE*YSIZE);
int size_Up = XSIZE*YSIZE;
int rows_shared = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gpu_mvprod<<<gridBlock,threadBlock>>>(x,y,alpha,NumUp,NumDown,dim,Umat,Down_data,Down_ind,size_Down,Up_data,Up_ind,size_Up,rows_shared);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
gpu_mvprod<<<gridBlock,threadBlock>>>(x,y,alpha,NumUp,NumDown,dim,Umat,Down_data,Down_ind,size_Down,Up_data,Up_ind,size_Up,rows_shared);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
gpu_mvprod<<<gridBlock,threadBlock>>>(x,y,alpha,NumUp,NumDown,dim,Umat,Down_data,Down_ind,size_Down,Up_data,Up_ind,size_Up,rows_shared);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}