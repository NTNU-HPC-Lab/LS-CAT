#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned int*) end_ptr) {
return;
}

unsigned int k = offset;
unsigned pat = pattern;
for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i++){
ptr[i] = pat;
k++;
if (k >= 32){
k=0;
pat = lb;
}else{
pat = pat << 1;
pat |= sval;
}
}

return;
}