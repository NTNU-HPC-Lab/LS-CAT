#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 16


__global__ void hello()
{
printf("Hello world! I'm a thread %d in block %d\n", threadIdx.x, blockIdx.x);
}