#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionParallel(unsigned char* image, unsigned char* new_image, unsigned height, unsigned width, int thread_count, int convolution_size)
{
// process image
int offset = (blockIdx.x * blockDim.x + threadIdx.x);
int width_out = (width - convolution_size + 1);
int height_out = (height - convolution_size + 1);

//Loop over pixels of smaller image
for (int i = offset; i < width_out * height_out * 4; i += thread_count)
{
int row = i / (4*width_out);
int col = i % (4*width_out);
int reference_pixel_offset = 4 * row * width + col;
float sum = 0.0;

if (convolution_size == 3)
{
float w[9] =
{
1,	2,		-1,
2,	0.25,	-2,
1,	-2,		-1
};

for (int j = 0; j < convolution_size; j++)
for (int k = 0; k < convolution_size; k++)
sum += image[reference_pixel_offset + 4 * k + 4 * j * width] * w[j * convolution_size + k];
}

if (convolution_size == 5)
{
float w[25] =
{
0.5,	0.75,	1,		-0.75,	-0.5,
0.75,	1,		2,		-1,		-0.75,
1,		2,		0.25,	-2,		-1,
0.75,	1,		-2,		-1,		-0.75,
0.5,	0.75,	-1,		-0.75,	-0.5
};

for (int j = 0; j < convolution_size; j++)
for (int k = 0; k < convolution_size; k++)
sum += image[reference_pixel_offset + 4 * k + 4 * j * width] * w[j * convolution_size + k];

}
if (convolution_size == 7)
{
float w[49] =
{
0.25,	0.3, 	0.5, 	0.75, 	-0.5, 	-0.3, 	-0.25,
0.3,	0.5,	0.75,	1,		-0.75,	-0.5, 	-0.3,
0.5,	0.75,	1,		2,		-1,		-0.75,	-0.5,
0.75,	1,		2,		0.25,	-2,		-1, 	-0.75,
0.5,	0.75,	1,		-2,		-1,		-0.75, 	-0.5,
0.3,	0.5,	0.75,	-1,		-0.75,	-0.5, 	-0.3,
0.25, 	0.3,	0.5,	-0.75,	-0.5, 	-0.3, 	-0.25

};

for (int j = 0; j < convolution_size; j++)
for (int k = 0; k < convolution_size; k++)
sum += image[reference_pixel_offset + 4 * k + 4 * j * width] * w[j * convolution_size + k];
}

if (sum <= 0)			sum = 0;
if (sum >= 255)			sum = 255;
if ((i + 1) % 4 == 0)	sum = 255; // Set a = 255

new_image[i] = (int) sum;

}

}