#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pool(unsigned char* image, unsigned char* new_image, unsigned height, unsigned width, int thread_count)
{
// process image
int offset = (blockIdx.x * blockDim.x + threadIdx.x)*4;

for (int i = offset; i < (width*height); i+=(thread_count*4) )
{
int x = i % (width * 2) * 2;
int y = i / (width * 2);
int p1 = 8 * width * y + x;
int p2 = 8 * width * y + x + 4;
int p3 = 8 * width * y + x + 4 *  width;
int p4 = 8 * width * y + x + 4 * width + 4;

unsigned r[] = { image[p1],   image[p2],   image[p3],   image[p4] };
unsigned g[] = { image[p1+1], image[p2+1], image[p3+1], image[p4+1] };
unsigned b[] = { image[p1+2], image[p2+2], image[p3+2], image[p4+2] };
unsigned a[] = { image[p1+3], image[p2+3], image[p3+3], image[p4+3] };

int rMax = r[0];
int gMax = g[0];
int bMax = b[0];
int aMax = a[0];

for (int j = 1; j < 4; j++ )
{
if (r[j] > rMax) rMax = r[j];
if (g[j] > gMax) gMax = g[j];
if (b[j] > bMax) bMax = b[j];
if (a[j] > aMax) aMax = a[j];

}
new_image[i] = rMax;
new_image[i+1] = gMax;
new_image[i+2] = bMax;
new_image[i+3] = aMax;


}
}