#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectify(unsigned char* image, unsigned height, unsigned width, int thread_count)
{
// process image
int block = (height * width * 4) / thread_count;
int offset = threadIdx.x * block;
for (int i = 0; i < block; i++)
{
int j = offset + i;
if (image[j] < 127)	image[j] = 127;

}

}