#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

const double TOLERANCE = 1.0e-10;

/*
cgsolver with CUDA support solves the linear equation A*x = b where A is of size m x n
*/

__global__ void mvm_gpu(double *A_cuda, double *X_cuda, double *Y_cuda, int *m_locals_cuda, int *A_all_pos_cuda, int n, int nthreads){
int t = blockIdx.x * blockDim.x + threadIdx.x;

if (t < nthreads){
for (int i=A_all_pos_cuda[t]; i<A_all_pos_cuda[t]+m_locals_cuda[t]; ++i) {
Y_cuda[i] = 0.;
for (int j=0; j<n; ++j)
Y_cuda[i] += A_cuda[i * n + j] * X_cuda[j];
}
}
}