#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>

//#include "cub/cub/cub.cuh"

namespace gpu_mg {

__global__ void init_vertex_group(int *row_group, bool *dl_matrix,
                                  int *vertex_num, int *t_cn, int *t_rn,
                                  int *offset_row, int *offset_matrix,
                                  int graph_count);

__global__ void
mc_solver(bool *dl_matrix, bool *transpose_dl_matrix, int *next_col,
          int *next_row, int *results, int *deleted_cols, int *deleted_rows,
          int *col_group, int *row_group, int *conflict_count, int *vertex_num,
          int *total_dl_matrix_row_num, int *total_dl_matrix_col_num,
          int *offset_col, int *offset_row, int *offset_matrix,
          int *search_depth, int *selected_row_id, int *current_conflict_count,
          int *conflict_node_id, int *conflict_col_id,
          int *existance_of_candidate_rows, int *conflict_edge, int *max,
          const int graph_count, const int hard_conflict_threshold,
          const int graph_per_block);
// void mc_solver(int* dl_matrix, int* results, int* deleted_cols, int*
// deleted_rows, int* col_group,int* row_group, int* conflict_count,	const
// int vertex_num, const int total_dl_matrix_row_num, const int
// total_dl_matrix_col_num);

} // namespace gpu_mg
