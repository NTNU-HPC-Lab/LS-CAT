#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void delete_rows_and_columns( bool *dl_matrix, bool *transpose_dl_matrix, const int *next_row, int *next_col, short *deleted_rows, short *deleted_cols, const int search_depth, const int selected_row_id, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
bool *selected_row = dl_matrix + selected_row_id * total_dl_matrix_col_num;
///*
for (int i = threadIdx.x; i < total_dl_matrix_col_num;
// // The below line will have negative effect of the col number is small
//  i += (next_col[selected_row_idx + i] + blockDim.x - 1) / blockDim.x
i += blockDim.x) {
if (deleted_cols[i] == 0 && selected_row[i] == 1) {
deleted_cols[i] = search_depth;
// atomicInc(&tmp_deleted_cols_count)
const bool *transpose_dl_matrix_ptr =
transpose_dl_matrix + i * total_dl_matrix_row_num;
int nr = next_row[i * total_dl_matrix_row_num];
for (int j = 0; j < total_dl_matrix_row_num;
j += nr, transpose_dl_matrix_ptr += nr) {
nr = next_row[i * total_dl_matrix_row_num + j];
if (deleted_rows[j] == 0 && *transpose_dl_matrix_ptr == 1
// dl_matrix[j * total_dl_matrix_col_num + i] == 1
) {
deleted_rows[j] = search_depth;
}
}
}
}
//*/
/*
int * tmp_row;
int * tmp_next_col;
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x){
tmp_row = dl_matrix + i * total_dl_matrix_col_num;
tmp_next_col = next_col + i * total_dl_matrix_col_num;
for (int j = 0; j < total_dl_matrix_col_num; j += tmp_next_col[j]){
if (tmp_row[j] + selected_row[j] == 2 && deleted_cols[j] !=-1){
deleted_rows[i] = deleted_rows[i]==0?search_depth:deleted_rows[i];
deleted_cols[j] = deleted_cols[j]==0?search_depth:deleted_cols[j];
}
}
}
*/
}
__global__ void delete_rows_and_columns(int *dl_matrix, int *deleted_rows, int *deleted_cols, const int search_depth, const int selected_row_id, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
if (dl_matrix[selected_row_id * total_dl_matrix_col_num + i] == 1 &&
deleted_cols[i] == 0) {
deleted_cols[i] = search_depth;
for (int j = 0; j < total_dl_matrix_row_num; j++) {
if (dl_matrix[j * total_dl_matrix_col_num + i] == 1 &&
deleted_rows[j] == 0) {
atomicExch(deleted_rows + j, search_depth);
}
}
}
}
}