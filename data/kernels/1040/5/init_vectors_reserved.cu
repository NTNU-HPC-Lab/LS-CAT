#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void init_vectors_reserved(short *vec, const int vec_length) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
if (vec[i] != -1) {
vec[i] = 0;
}
}
}
__device__ void init_vectors(short *vec, const int vec_length) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
vec[i] = 0;
}
}
__global__ void init_vectors_reserved(int *vec, const int vec_length) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
if (vec[i] != -1) {
vec[i] = 0;
}
}
}