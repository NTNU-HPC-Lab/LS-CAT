#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void recover_deleted_cols(short *deleted_cols, const int search_depth, const int total_dl_matrix_col_num) {
for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
if (deleted_cols[i] >= search_depth) {
deleted_cols[i] = 0;
}
}
}
__global__ void recover_deleted_cols(int *deleted_cols, const int search_depth, const int total_dl_matrix_col_num) {
for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
if (deleted_cols[i] >= search_depth) {
deleted_cols[i] = 0;
}
}
}