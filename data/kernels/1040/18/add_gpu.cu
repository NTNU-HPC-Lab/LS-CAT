#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void add_gpu(int *device_var, int val) {
atomicAdd(device_var, val);
}
__global__ void add_gpu(int *device_arr, int device_idx, int val) {
device_arr[device_idx] += val;
//atomicAdd(&(device_arr[*device_idx]), val);
}