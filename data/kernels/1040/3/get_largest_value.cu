#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_largest_value(short *vec, const int vec_length, int *max) {

for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {

atomicMax(max, vec[i]);
}
}
__global__ void get_largest_value(int *vec, const int vec_length, int* max) {

for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
atomicMax(max, vec[i]);
}

}