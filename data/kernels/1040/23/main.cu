#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "init_vertex_group.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *row_group = NULL;
hipMalloc(&row_group, XSIZE*YSIZE);
bool *dl_matrix = NULL;
hipMalloc(&dl_matrix, XSIZE*YSIZE);
int *vertex_num = NULL;
hipMalloc(&vertex_num, XSIZE*YSIZE);
int *t_cn = NULL;
hipMalloc(&t_cn, XSIZE*YSIZE);
int *t_rn = NULL;
hipMalloc(&t_rn, XSIZE*YSIZE);
int *offset_row = NULL;
hipMalloc(&offset_row, XSIZE*YSIZE);
int *offset_matrix = NULL;
hipMalloc(&offset_matrix, XSIZE*YSIZE);
int graph_count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
init_vertex_group<<<gridBlock,threadBlock>>>(row_group,dl_matrix,vertex_num,t_cn,t_rn,offset_row,offset_matrix,graph_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
init_vertex_group<<<gridBlock,threadBlock>>>(row_group,dl_matrix,vertex_num,t_cn,t_rn,offset_row,offset_matrix,graph_count);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
init_vertex_group<<<gridBlock,threadBlock>>>(row_group,dl_matrix,vertex_num,t_cn,t_rn,offset_row,offset_matrix,graph_count);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}