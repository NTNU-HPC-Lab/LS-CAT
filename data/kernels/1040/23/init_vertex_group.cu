#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_vertex_row_group(int *row_group, bool *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
// printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
for (int j = 0, cur_index = i * total_dl_matrix_col_num; j < vertex_num;
j++, cur_index++) {
row_group[i] += (int)(dl_matrix[cur_index]) * (j + 1);
}
}
}
__global__ void init_vertex_group(int *row_group, bool *dl_matrix, int *vertex_num, int *t_cn, int *t_rn, int *offset_row, int *offset_matrix, int graph_count) {
int k = blockIdx.x;
if (k < graph_count) {
get_vertex_row_group(row_group + offset_row[k],
dl_matrix + offset_matrix[k], vertex_num[k], t_rn[k],
t_cn[k]);
}
}