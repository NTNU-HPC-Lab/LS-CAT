#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void recover_results(short *results, const int search_depth, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (results[i] == search_depth) {
results[i] = 0;
}
}
}
__global__ void recover_results(int *results, const int search_depth, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (results[i] == search_depth) {
results[i] = 0;
}
}
}