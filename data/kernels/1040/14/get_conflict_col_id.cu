#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_conflict_col_id(bool *dl_matrix, short *deleted_cols, int *conflict_col_id, int *conflict_edge, int total_dl_matrix_col_num, int vertex_num) {
// if(threadIdx.x==0){
//  printf("conflict edge a %d edge b
//  %d\n",conflict_edge[0],conflict_edge[1]);
// }
bool *edge_a_dlmatrix =
dl_matrix + conflict_edge[0] * total_dl_matrix_col_num;
bool *edge_b_dlmatrix =
dl_matrix + conflict_edge[1] * total_dl_matrix_col_num;
for (int j = threadIdx.x; j < total_dl_matrix_col_num; j = j + blockDim.x) {
if (edge_a_dlmatrix[j] == edge_b_dlmatrix[j] && deleted_cols[j] > 0 &&
edge_b_dlmatrix[j] == 1) {
atomicMax(conflict_col_id, j);
}
}
}
__global__ void get_conflict_col_id(int *dl_matrix, int *deleted_cols, int *conflict_col_id, int *conflict_edge, int total_dl_matrix_col_num, int vertex_num){
//if(threadIdx.x==0){
//  printf("conflict edge a %d edge b %d\n",conflict_edge[0],conflict_edge[1]);
// }
for (int j = threadIdx.x; j < total_dl_matrix_col_num;
j = j + blockDim.x) {
if (dl_matrix[conflict_edge[0] * total_dl_matrix_col_num + j]
== dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j] &&
deleted_cols[j] > 0 && dl_matrix[conflict_edge[1] * total_dl_matrix_col_num + j]==1) {
atomicMax(conflict_col_id, j);
}
}
__syncthreads();
}