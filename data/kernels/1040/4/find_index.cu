#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void find_index(short *vec, const int vec_length, int *value, int *index) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
if (vec[i] == *value) {
atomicMax(index, i);
}
}
}
__global__ void find_index(int *vec, const int vec_length, int *value, int *index){
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
if(vec[i]==*value){
atomicMax(index, i);
}

}
}