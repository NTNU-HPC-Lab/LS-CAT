#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_vertex_row_group(int *row_group, bool *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
// printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
for (int j = 0, cur_index = i * total_dl_matrix_col_num; j < vertex_num;
j++, cur_index++) {
row_group[i] += (int)(dl_matrix[cur_index]) * (j + 1);
}
}
}
__global__ void get_vertex_row_group(int *row_group, int *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
// printf("%d %d\n", vertex_num, total_dl_matrix_row_num);
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
for (int j = 0; j < vertex_num; j++) {
row_group[i] += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
}
}
}