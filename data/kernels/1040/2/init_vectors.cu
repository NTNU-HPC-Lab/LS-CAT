#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void init_vectors(short *vec, const int vec_length) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
vec[i] = 0;
}
}
__global__ void init_vectors(int *vec, const int vec_length) {
for (int i = threadIdx.x; i < vec_length; i = i + blockDim.x) {
vec[i] = 0;
}
}