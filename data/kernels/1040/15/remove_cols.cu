#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void remove_cols(short *deleted_cols, int *col_group, const int conflict_col_id, const int total_dl_matrix_col_num) {
for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
if (col_group[i] == col_group[conflict_col_id]) {
deleted_cols[i] = -1;
}
}
}
__global__ void remove_cols(int *deleted_cols, int *col_group, const int conflict_col_id, const int total_dl_matrix_col_num) {
for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
if (col_group[i] == col_group[conflict_col_id]) {
deleted_cols[i] = -1;
}
}
}