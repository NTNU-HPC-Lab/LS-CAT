#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void check_existance_of_candidate_rows( short *deleted_rows, int *row_group, const int search_depth, int *token, int *selected_row_id, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
// std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
// std::cout<<"Candidate Row Found...."<<std::endl;
// atomicExch(token, 1);
*token = 1;
atomicMin(selected_row_id, i);
// If find a number can break;
// break;
}
}
}
__global__ void check_existance_of_candidate_rows( int *deleted_rows, int *row_group, const int search_depth, int *token, int *selected_row_id, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
// std::cout<<deleted_rows[i]<<' '<<row_group[i]<<std::endl;
if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
// std::cout<<"Candidate Row Found...."<<std::endl;
atomicExch(token, 1);
atomicMin(selected_row_id, i);
}
}
__syncthreads();
}