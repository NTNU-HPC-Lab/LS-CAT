#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_conflict_edge(bool *dl_matrix, short *deleted_rows, int *row_group, const int conflict_node_id, const int search_depth, int *conflict_edge, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
//*conflict_col_id = 0;
// int idxa = 0;
// int idxb = 0;

for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
// find the conflict edge that connects current node and the most closest
// node.
if (deleted_rows[i] == -conflict_node_id) {
atomicMax(conflict_edge, i);
}
if (row_group[i] == search_depth + 1 &&
deleted_rows[i] == conflict_node_id) {
atomicMax(conflict_edge + 1, i);
}
}
}
__global__ void get_conflict_edge(int *dl_matrix, int *deleted_rows, int *deleted_cols, int *row_group, const int conflict_node_id, const int search_depth, int *conflict_edge, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
//*conflict_col_id = 0;
//int idxa = 0;
//int idxb = 0;

for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
// find the conflict edge that connects current node and the most closest
// node.
if (deleted_rows[i] == -conflict_node_id) {
atomicMax(conflict_edge, i);
}
if (row_group[i] == search_depth + 1 &&
deleted_rows[i] == conflict_node_id) {
atomicMax(conflict_edge+1, i);
}
}
__syncthreads();
}