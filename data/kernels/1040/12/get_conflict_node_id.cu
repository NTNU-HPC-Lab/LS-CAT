#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void get_conflict_node_id(short *deleted_rows, int *row_group, const int search_depth, int *conflict_node_id, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (row_group[i] == search_depth + 1 &&
deleted_rows[i] < search_depth + 1) {
atomicMax(conflict_node_id, deleted_rows[i]);
}
}
}
__global__ void get_conflict_node_id(int *deleted_rows, int *row_group, const int search_depth, int *conflict_node_id, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (row_group[i] == search_depth + 1 &&  deleted_rows[i] < search_depth+1) {
atomicMax(conflict_node_id, deleted_rows[i]);
}
}
__syncthreads();
}