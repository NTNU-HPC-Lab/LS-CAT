#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void recover_deleted_rows(short *deleted_rows, const int search_depth, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (abs(deleted_rows[i]) > search_depth ||
deleted_rows[i] == search_depth) {
deleted_rows[i] = 0;
}
}
}
__global__ void recover_deleted_rows(int *deleted_rows, const int search_depth, const int total_dl_matrix_row_num) {
for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
if (abs(deleted_rows[i]) > search_depth ||
deleted_rows[i] == search_depth) {
deleted_rows[i] = 0;
}
}
}