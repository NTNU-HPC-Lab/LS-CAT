#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned char value(float n1, float n2, int hue) {
if (hue > 360) hue -= 360;
else if (hue < 0) hue += 360;

if (hue < 60)
return (unsigned char) (255 * (n1 + (n2 - n1) * hue / 60));
if (hue < 180)
return (unsigned char) (255 * n2);
if (hue < 240)
return (unsigned char) (255 * (n1 + (n2 - n1) * (240 - hue) / 60));
return (unsigned char) (255 * n1);
}
__global__ void randomize(float* array, hiprandState* rand, unsigned long N)
{
int x = threadIdx.x + (blockIdx.x * blockDim.x);
int y = threadIdx.y + (blockIdx.y * blockDim.y);
unsigned long tid = x + (y * blockDim.x * gridDim.x);

if(tid < N){
hiprandState localState = rand[tid]; // get local hiprandState as seed
float theRand = hiprand_uniform(&localState); // use to get value from 0-1
rand[tid] = localState; // save new state as previous state for next gen

array[tid] = theRand;
}

}