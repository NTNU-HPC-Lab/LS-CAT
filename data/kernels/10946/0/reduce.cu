#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
double iStart1, iStart2, iStart3a, iStart3b, iStart4a, iStart4b, iStart4c, iStart5;
double iElaps1=0, iElaps2=0, iElaps3a=0, iElaps3b=0, iElaps4=0, iElaps5=0;
// Hold configurations for Kmeans
struct Info {
int     numPoints;
int     dim;
int     numCentroids;
int     numRepeats;
int    *belongs;
float **points;
float **centroids;
int     thresholdLoops;
float   thresholdFraction;
int     threadPerBlock;
};

// ************* Utils ************* //

__global__ static void reduce(int *g_idata, int l1, int l2) {
extern __shared__ unsigned int sdata[];
unsigned int tid = threadIdx.x;

if (tid < l1) {
sdata[tid] = g_idata[tid];
} else {
sdata[tid] = 0;
}
__syncthreads();

// Parallel Reduction (l2 must be power of 2)
for (unsigned int s = l2 / 2; s > 0; s >>= 1) {
if (tid < s)     {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

if (tid == 0) {
g_idata[0] = sdata[0];
}
}