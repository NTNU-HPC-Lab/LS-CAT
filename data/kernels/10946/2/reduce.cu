#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void reduce(int *g_idata, int l1, int l2) {
extern __shared__ unsigned int sdata[];
unsigned int tid = threadIdx.x;

if (tid < l1) {
sdata[tid] = g_idata[tid];
} else {
sdata[tid] = 0;
}
__syncthreads();

// Parallel Reduction (l2 must be power of 2)
for (unsigned int s = l2 / 2; s > 0; s >>= 1) {
if (tid < s)     {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

if (tid == 0) {
g_idata[0] = sdata[0];
}
}