#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_Gamma_kernel(double* Gamma, int Gamma_n, int Gamma_ld, double* N, int N_r, int N_c, int N_ld, double* G, int G_r, int G_c, int G_ld, int* random_vertex_vector, double* exp_V, double* exp_delta_V) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int vertex_index = N_c - G_c;

if (i < Gamma_n and j < Gamma_n) {
int configuration_e_spin_index_i = random_vertex_vector[i];
int configuration_e_spin_index_j = random_vertex_vector[j];

if (configuration_e_spin_index_j < vertex_index) {
double delta = 0;

if (configuration_e_spin_index_i == configuration_e_spin_index_j)
delta = 1.;

double N_ij = N[configuration_e_spin_index_i + configuration_e_spin_index_j * N_ld];

Gamma[i + j * Gamma_ld] = (N_ij * exp_V[j] - delta) / (exp_V[j] - 1.);
}
else
Gamma[i + j * Gamma_ld] =
G[configuration_e_spin_index_i + (configuration_e_spin_index_j - vertex_index) * G_ld];
}

if (i < Gamma_n and j < Gamma_n and i == j) {
double gamma_k = exp_delta_V[j];
Gamma[i + j * Gamma_ld] -= (gamma_k) / (gamma_k - 1.);
}
}