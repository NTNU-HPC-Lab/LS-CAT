#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_row_on_Gamma_matrix_kernel(int row_index, int vertex_index, int* indices, double* exp_V, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* row_ptr, int incr) {
// int l = threadIdx.x;
int l = blockIdx.x;

int i_index, j_index;
double delta;

i_index = indices[row_index];
j_index = indices[l];

if (j_index < vertex_index) {
delta = i_index == j_index ? 1 : 0;
row_ptr[l * incr] = (N_ptr[i_index + LD_N * j_index] * exp_V[l] - delta) / (exp_V[l] - 1.);
}
else
row_ptr[l * incr] = G_ptr[i_index + LD_G * (j_index - vertex_index)];
}