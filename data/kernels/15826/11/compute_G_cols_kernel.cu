#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_G_cols_kernel(int N_i, int N_r, int N_c, int* p_ptr, double* exp_V_ptr, double* N_ptr, int N_ld, double* G_ptr, int G_ld, double* G_cols_ptr, int G_cols_ld) {
int I = threadIdx.x + blockIdx.x * BLOCK_SIZE_x;  // blockDim.x;

int l_MIN = BLOCK_SIZE_y * (blockIdx.y + 0);
int l_MAX = BLOCK_SIZE_y * (blockIdx.y + 1);

l_MIN = max(l_MIN, 0);
l_MAX = min(l_MAX, N_i);

if (I < N_r) {
// for(int l=0; l<N_i; ++l)
for (int l = l_MIN; l < l_MAX; ++l) {
if (p_ptr[l] >= N_c) {
G_cols_ptr[I + l * G_cols_ld] = G_ptr[I + (p_ptr[l] - N_c) * G_ld];
}
else {
double alpha = exp_V_ptr[l] / (exp_V_ptr[l] - 1.);

G_cols_ptr[I + l * G_cols_ld] = alpha * N_ptr[I + p_ptr[l] * N_ld];
}
}

// for(int l=0; l<N_i; ++l)
for (int l = l_MIN; l < l_MAX; ++l)
if (p_ptr[l] < N_c and I == p_ptr[l])
G_cols_ptr[I + l * G_cols_ld] -= 1. / (exp_V_ptr[l] - 1.);
}
}