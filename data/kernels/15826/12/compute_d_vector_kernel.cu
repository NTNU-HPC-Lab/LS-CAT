#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_d_vector_kernel(int N_i, int* d_ind, double* d_ptr, int* p_ptr, double* N_ptr, int N_ld) {
int I = threadIdx.x + blockIdx.x * blockDim.x;

if (I < N_i) {
int index = p_ptr[d_ind[I]];

d_ptr[d_ind[I]] = 1. / N_ptr[index + index * N_ld];
}
}