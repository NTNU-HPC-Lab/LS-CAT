#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA: grid stride looping
#ifndef CUDA_KERNEL_LOOP
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
#endif





__global__ void psamask_collect_forward_cuda(const int nthreads, const int feature_H_, const int feature_W_, const int mask_H_, const int mask_W_, const int half_mask_H_, const int half_mask_W_, const float* mask_data, float* buffer_data) {
CUDA_KERNEL_LOOP(index, nthreads) {
const int w = index % feature_W_;
const int h = (index / feature_W_) % feature_H_;
const int n = index / feature_W_ / feature_H_;
// effective mask region : [hstart, hend) x [wstart, wend) with mask-indexed
const int hstart = max(0, half_mask_H_ - h);
const int hend = min(mask_H_, feature_H_ + half_mask_H_ - h);
const int wstart = max(0, half_mask_W_ - w);
const int wend = min(mask_W_, feature_W_ + half_mask_W_ - w);
// (hidx,                    widx                   ) with mask-indexed
// (hidx + h - half_mask_H_, widx + w - half_mask_W_) with feature-indexed
for (int hidx = hstart; hidx < hend; hidx++) {
for (int widx = wstart; widx < wend; widx++) {
buffer_data[(n * feature_H_ * feature_W_ + (hidx + h - half_mask_H_) * feature_W_ + (widx + w - half_mask_W_)) * feature_H_ * feature_W_ + h * feature_W_ + w] =
mask_data[((n * mask_H_ * mask_W_ + hidx * mask_W_ + widx) * feature_H_ + h) * feature_W_ + w];
}
}
}
}