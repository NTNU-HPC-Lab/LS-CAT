#include "hip/hip_runtime.h"
#include "includes.h"

// Lets you use the Cuda FFT library



hipError_t mathWithCuda(float *output, float *input1, float *input2, unsigned int size, int oper);

// Using __global__ to declare function as device code (GPU)
// Do the math inside here:

// Helper function for using CUDA to add vectors in parallel.
__global__ void mathKernel(float *output, float *input1, float *input2, int n, int oper)
{
// Allocate elements to threads
int i = threadIdx.x + blockIdx.x * blockDim.x;

// Avoid access beyond the end of the array
if (i < n)
{
// No for-loop needed, CUDA runtime will thread this
switch (oper)
{
case 1: // Addition
output[i] = input1[i] + input2[i];
break;
case 2: // Subtraction
output[i] = input1[i] - input2[i];
break;
case 3: // Multiplication
output[i] = input1[i] * input2[i];
break;
case 4: // Division
output[i] = input1[i] / input2[i];
break;

// Add more operations here:
case 5:
break;
case 6:
break;
case 7:
break;

default:
return;
}

// Ensure all the data is available
__syncthreads(); // Gives a syntax "error" but this doesn't give build errors
}
}