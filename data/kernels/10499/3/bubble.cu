#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void bubble(unsigned int length, unsigned int parity, float* tab)
{

int index = 2* (threadIdx.x + blockDim.x * blockIdx.x);
int  leftElementID = index + parity;
int rightElementID = index + parity + 1;

float l, r;
if (rightElementID < length)
{
l = tab[  leftElementID ];
r = tab[ rightElementID ];
if ( r < l )
{
tab[  leftElementID ] = r;
tab[ rightElementID ] = l;
}
}


}