#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add(int n, float *a, float *sum)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i<n)
{
for (int j = 0; j < n; j++)
{
sum[i] = sum[i] + a[i*n + j];
}
}

}