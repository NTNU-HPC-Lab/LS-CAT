#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void wavee(int* tab, unsigned int rowSize, unsigned int centerX, unsigned int centerY, float A, float lambda, float time, float fi, unsigned int N)
{

int index = threadIdx.x + blockDim.x * blockIdx.x;
int w = int(index/rowSize);
int h = index%rowSize;

if ( w*rowSize+h < N ) {
float dx = 0;
if(centerX > w) {
dx = centerX - w;
} else {
dx = w - centerX;
}
float dy = 0;
if(centerY > h) {
dy = centerY - h;
} else {
dy = h - centerY;
}
float distance = pow(dx,2) + pow(dy,2);
distance = sqrt(distance);

float pi = 3.1415f;
float v = 1.0f;
float T = lambda/v;
float ww = 2.0f*pi/T;
float k = 2.0f*pi/lambda;
float f = A * sin( ww*time - k*distance + fi );

float res = f * 127 + 127;
tab[index] = int(res);

}


}