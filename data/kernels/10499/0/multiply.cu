#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void multiply(int n, float *a, float *b, float *sum)
{
int ind = threadIdx.x + blockDim.x * blockIdx.x;
int i = ind ;
int j = ind % n;

if (j<n)
{
sum[i] = a[i] * b[j];
}

}