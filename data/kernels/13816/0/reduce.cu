#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *in, int *out, int N) {
int sum = 0;
for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < N;
i += blockDim.x * gridDim.x) {
sum += in[i];
}
//sum = warpReduceSum(sum);
//if (threadIdx.x & (warpSize - 1) == 0) atomicAdd(out, sum);
}