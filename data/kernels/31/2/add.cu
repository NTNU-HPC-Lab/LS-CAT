#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( float *x, float *y, float *z, float *deltaX, float *deltaY, float *deltaZ ) {
int tid = blockIdx.x;    // this thread handles the data at its thread id
if (tid < N)
x[tid] = x[tid] + deltaX[tid];
if (tid < N)
y[tid] = y[tid] + deltaY[tid];
if (tid<N)
z[tid] = z[tid] + deltaZ[tid];

}