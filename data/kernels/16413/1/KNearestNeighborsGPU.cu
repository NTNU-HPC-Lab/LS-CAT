#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt);

// cpu algorithm
__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= cnt) return;

float3 point = dataArray[id], current;
float minimumDist = 3.4028234664e38f, distance = 0;

for (int j = 0; j < cnt; j++)
{
if (id == j) continue;

current = dataArray[j];
distance = (point.x - current.x) * (point.x - current.x);
distance += (point.y - current.y) * (point.y - current.y);
distance += (point.z - current.z) * (point.z - current.z);

if (distance < minimumDist)
{
minimumDist = distance;
result[id] = j;
}
}
}