#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_standarization(float *data, int rows, int columns) {
int total_threads_count = blockDim.x * gridDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
float var, ave, amo;

for (int i = tid+1; i < columns; i=i+total_threads_count) {
amo = 0, var = 0;
for (int j = 0; j < rows; ++j) {
amo = amo + *(data + (j * columns) + i);
}
ave  = amo / float(rows);

for (int j = 0; j < rows; ++j) {
float factor = *(data + (j * columns) + i) - ave;
var = var + (factor * factor);
}

if (var == 0) {
for (int j = 0; j < rows; j++) {
*(data + (j * columns) + i) = *(data + (j * columns) + i) / 255.;
}
continue;
}

float sd_reciprocal = 1./sqrt(var);

for (int j = 0; j < rows; j++) {
*(data + (j * columns) + i) = (*(data + (j * columns) + i) - ave) * sd_reciprocal;
}
}
}