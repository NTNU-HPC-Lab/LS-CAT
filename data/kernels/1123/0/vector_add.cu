#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000
#define MAX_ERR 1e-6


__global__ void vector_add(float *out, float *a, float *b, int n) {
for(int i = 0; i < n; i ++){
out[i] = a[i] + b[i];
}
}