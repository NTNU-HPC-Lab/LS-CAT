#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stdp_kernel( float *weight, int weight_size_0, int weight_size_1, int weight_size_2, int weight_size_3, float *output_spike, int output_spike_size_0, int output_spike_size_1, int output_spike_size_2, int output_spike_size_3, float *history, float *weight_update)
{

// Each thread is for an element in output_spike in the form:
// (batch_id, channel_id, height_id, width_id) = (blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x).
// Note that grid is 3D and block is 1D; this to make the channel the fastest dimension as across
// the channel the history patch block is accessed is same.
// IMPORTANT: Number of channels MUST BE LESS than maximum thread limit.
// Note: All 'id's refer to numpy indices and 'index'/'linear_idx' refer to linear memory index.

const int offset = weight_size_2 / 2;
const int batch_id = blockIdx.x;
const int channel_id = threadIdx.x;
const int height_id = blockIdx.y;
const int width_id = blockIdx.z;

/*
// some constants
output_spike_size_1_2_3 = output_spike_size_1 * output_spike_size_2 * output_spike_size_3;
output_spike_size_2_3 = output_spike_size_2 * output_spike_size_3;
output_spike_size_3 = output_spike_size_3; // = gridDim.z;
weight_size_0_1_2_3 = weight_size_0 * weight_size_1 * weight_size_2 * weight_size_3;
weight_size_1_2_3 = weight_size_1 * weight_size_2 * weight_size_3;
weight_size_2_3 = weight_size_2 * weight_size_3;
weight_size_1 = weight_size_1
weight_size_3 = weight_size_3
*/

const int linear_idx = (batch_id * (output_spike_size_1 * output_spike_size_2 * output_spike_size_3)) \
+ (channel_id * (output_spike_size_2 * output_spike_size_3)) \
+ (height_id * (output_spike_size_3)) \
+ width_id;

if (output_spike[linear_idx] != 1.0f)
return;

const int filter_index = channel_id * (weight_size_1 * weight_size_2 * weight_size_3);
const int delta_index_const = batch_id * (weight_size_0 * weight_size_1 * weight_size_2 * weight_size_3);

float input_tmp;
int w_index, deltaW_index;
int filter_2d_index, filter_1d_index;
int history_1d_index, history_pivot, history_index, history_index1, history_index2;

for (int l = 0; l < weight_size_1; l++)
{
history_pivot = (batch_id * (weight_size_1 * output_spike_size_2 * output_spike_size_3)) \
+ (l * (output_spike_size_2 * output_spike_size_3));
filter_2d_index = filter_index + (l * weight_size_2 * weight_size_3);

for (int i = 0; i < weight_size_2; i++)
{
filter_1d_index = filter_2d_index + (i * weight_size_3);
history_index1 = height_id + i - offset;
if (history_index1 < 0 || history_index1 >= output_spike_size_2)
continue;
history_1d_index = history_pivot + (history_index1 * output_spike_size_3);

for (int j = 0; j < weight_size_3; j++)
{
w_index = filter_1d_index + j;
deltaW_index = delta_index_const + w_index;
history_index2 = width_id + j - offset;
if (history_index2 < 0 || history_index2 >= output_spike_size_3)
continue;
history_index = history_1d_index + history_index2;
input_tmp = history[history_index];
//weight_update[deltaW_index] = deltaW_index;
weight_update[deltaW_index] = (weight[w_index] * (1-weight[w_index])) \
* ((input_tmp != 0.0f) - (input_tmp == 0.0f));
}
}
}
}