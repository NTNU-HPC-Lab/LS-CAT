#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void processBoxes(int size, const float* src, float* dst,const int stridex, const int stridey)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < size)
{
float4* src_boxes = (float4*)src + index;
float4* dst_boxes = (float4*)dst + index;
float4 boxes = *src_boxes;
float4 new_boxes = {0};
new_boxes.x = boxes.x - boxes.z * stridex / 2;
new_boxes.y = boxes.y - boxes.w * stridey / 2;
new_boxes.z = boxes.x + boxes.z * stridex / 2;
new_boxes.w = boxes.y + boxes.w * stridey / 2;
*dst_boxes = new_boxes;
}
}