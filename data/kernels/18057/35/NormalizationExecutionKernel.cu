#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizationExecutionKernel(unsigned char* src, float* dst, const int size, const float alpha, const float beta, const float bias)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < size){
dst[index] = (float)(src[index] - alpha) / beta + bias;
}
}