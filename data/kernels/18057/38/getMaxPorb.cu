#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getMaxPorb(const int size, const float* class_prob, const int class_num, float* max_prob, int* idx, int *class_idx, const int conf_thresh)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < size)
{
// printf("run here %d!\n", index);
float temp_max_prob = 0.0f;
const float *start = class_prob + index * class_num;
int class_index = -1;
for(int i = 0; i < class_num; i++)
{
float curr_prob = start[i];
if(temp_max_prob <= curr_prob)
{
class_index = i;
temp_max_prob = curr_prob;
}
}
max_prob[index] = 0.0f;
if(temp_max_prob >= conf_thresh)
{
// atomicAdd(detecNum, 1);
max_prob[index] = temp_max_prob;
// printf("run here %d!\n", index);
}
idx[index] = index;
class_idx[index] = class_index;
}
}