#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

























































































































































































}
__global__ void uplo_inv_cbrt (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < sd);
const bool check = valid &&
((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
if (check) {
b[offset_b + gid_0 + gid_1 * ld_b] = CAST(rcbrt)(a[offset_a + gid_0 + gid_1 * ld_a]);
}
}