#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif





















}
__global__ void vector_axpby (const int n, const REAL alpha, const REAL* x, const int offset_x, const int stride_x, const REAL beta, REAL* y, int offset_y, int stride_y) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
const int ix = offset_x + gid * stride_x;
const int iy = offset_y + gid * stride_y;
y[iy] = alpha * x[ix] + beta * y [iy];
}
}