#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

























































































































































































}
__global__ void ge_sincos (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < fd);
if (valid) {
CAST(sincos)(a[offset_a + gid_0 + gid_1 * ld_a],
&b[offset_b + gid_0 + gid_1 * ld_b], &c[offset_c + gid_0 + gid_1 * ld_c]);
}
}