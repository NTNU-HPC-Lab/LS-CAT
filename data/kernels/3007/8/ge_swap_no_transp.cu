#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif





















}
__global__ void ge_swap_no_transp (const int sd, const int fd, REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < fd);
if (valid) {
const int ia = offset_a + gid_0 + gid_1 * ld_a;
const int ib = offset_b + gid_0 + gid_1 * ld_b;
const REAL c = b[ib];
b[ib] = a[ia];
a[ia] = c;
}
}