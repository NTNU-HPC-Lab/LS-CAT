#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif





}
__global__ void vector_copy (const int n, const NUMBER* x, const int offset_x, const int stride_x, NUMBER* y, const int offset_y, const int stride_y) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
const int ix = offset_x + gid * stride_x;
const int iy = offset_y + gid * stride_y;
y[iy] = x[ix];
}
}