#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif





















}
__global__ void ge_set (const int sd, const int fd, const REAL val, REAL* a, const int offset_a, const int ld_a) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < fd);
if (valid) {
a[offset_a + gid_0 + gid_1 * ld_a] = val;
}
}