#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif





}
__global__ void vector_set (const int n, const NUMBER val, NUMBER* x, const int offset_x, const int stride_x) {
const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
x[offset_x + gid * stride_x] = val;
}
}