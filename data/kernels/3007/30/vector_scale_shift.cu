#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

























































































































































































}
__global__ void vector_scale_shift (const int n, const REAL* x, const int offset_x, const int stride_x, const REAL scalea, const REAL shifta, const REAL scaleb, const REAL shiftb, REAL* y, const int offset_y, const int stride_y) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
y[offset_y + gid * stride_y] = scalea * x[offset_x + gid * stride_x] + shifta;

}
}