#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

























































































































































































}
__global__ void vector_pow (const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
z[offset_z + gid * stride_z] =
CAST(pow)(x[offset_x + gid * stride_x], y[offset_y + gid * stride_y]);
}
}