#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif





















}
__global__ void uplo_scal (const int sd, const int unit, const int bottom, const REAL alpha, REAL* a, const int offset_a, const int ld_a) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < sd);
const bool check = valid &&
((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
if (check) {
a[offset_a + gid_0 + gid_1 * ld_a] *= alpha;
}
}