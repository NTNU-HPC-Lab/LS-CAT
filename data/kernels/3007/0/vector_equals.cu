#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif





}
__global__ void vector_equals (const int n, const NUMBER* x, const int offset_x, const int stride_x, const NUMBER* y, const int offset_y, const int stride_y, int* eq_flag) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
const int ix = offset_x + gid * stride_x;
const int iy = offset_y + gid * stride_y;
if (x[ix] != y[iy]) {
eq_flag[0]++;
}
}
}