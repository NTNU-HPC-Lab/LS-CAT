#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

























































































































































































}
__global__ void vector_log1p (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

const int gid = blockIdx.x * blockDim.x + threadIdx.x;
if (gid < n) {
y[offset_y + gid * stride_y] = CAST(log1p)(x[offset_x + gid * stride_x]);
}
}