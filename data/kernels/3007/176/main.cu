#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "uplo_asin.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int sd = 1;
const int unit = 1;
const int bottom = 1;
const REAL *a = NULL;
hipMalloc(&a, XSIZE*YSIZE);
const int offset_a = 1;
const int ld_a = 1;
REAL *b = NULL;
hipMalloc(&b, XSIZE*YSIZE);
const int offset_b = 1;
const int ld_b = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
uplo_asin<<<gridBlock,threadBlock>>>(sd,unit,bottom,a,offset_a,ld_a,b,offset_b,ld_b);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
uplo_asin<<<gridBlock,threadBlock>>>(sd,unit,bottom,a,offset_a,ld_a,b,offset_b,ld_b);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
uplo_asin<<<gridBlock,threadBlock>>>(sd,unit,bottom,a,offset_a,ld_a,b,offset_b,ld_b);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}