#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif





















}
__global__ void uplo_copy_no_transp (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
const bool valid = (gid_0 < sd) && (gid_1 < sd);
const bool check = valid &&
((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
if (check) {
const int ia = offset_a + gid_0 + gid_1 * ld_a;
const int ib = offset_b + gid_0 + gid_1 * ld_b;
b[ib] = a[ia];
}
}