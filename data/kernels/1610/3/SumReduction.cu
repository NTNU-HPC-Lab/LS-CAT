#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumReduction(int* input, int n)
{
// Handle to thread block group
extern __shared__ int sm[];

// load shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

sm[tid] = (i < n) ? input[i] : 0;

__syncthreads();

// do reduction in shared mem
for (unsigned int s = 1; s < blockDim.x; s *= 2)
{
if ((tid % (2 * s)) == 0)
{
sm[tid] += sm[tid + s];
}

__syncthreads();
}

// write result for this block to global mem
//printf("%d: %d   , block ID: %d \n", threadIdx.x, sm[tid], blockIdx.x);
if (tid == 0) input[blockIdx.x] = sm[0];

}