#include "hip/hip_runtime.h"
#include "includes.h"
/*
There can be problem with crashing app
It is caused by WDDM TDR delay
this delay works in such a way that kill the kernel if it doesnt finish in specific time
so for big numbers it can be a problem
but you can change time or even turn it off in Nsight monitor : option->general->microsoft display driver
*/




#define PI 3.14159265358979323846


#define N	10000	//data size
#define ES	10000	//estimation size
#define HS	20		//histogram size	the lower hs is the better results will appear
//do not spoil and dont set data size greater than histogram size


__global__ void estimationKernel(float* data, size_t n, float* kernelEstimation, size_t es, float dx, float h)
{
int index = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = index; i < es; i += stride)
{
float di;
di = dx * i;	//on which position on OX axis we calculate the estimation

kernelEstimation[i] = 0;
for (int j = 0; j < n; j++)
{
//formula:
float power = -0.5f * (di - data[j]) * (di - data[j]) / h / h;
kernelEstimation[i] += expf(power);

}
kernelEstimation[i] /= (n * h) * sqrt(2 * PI);	//also formula

}
}