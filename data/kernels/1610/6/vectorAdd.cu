#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(int numElements, float *x, float *y)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
y[i] = x[i] + y[i];
}
}