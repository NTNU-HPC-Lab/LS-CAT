#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rearrangePopulationWithRange(float *gene, float *fit, int *range)
{
const int idx = threadIdx.x + blockDim.x*blockIdx.x;
if(range[0]>range[1]) return;

int totalElements = range[1] - range[0] + 1;
int nHalf = totalElements / 2;
if(idx> nHalf) return;

int i = range[0] + idx;
int j = range[1] - idx;

if (fit[i] < fit[j]) {
for(int k=0; k<6; k++) {
float t = gene[i*6+k];
gene[i*6+k] = gene[j*6+k];
gene[j*6+k] = t;

}
float t = fit[i];
fit[i] = fit[j];
fit[j] = t;
}
}