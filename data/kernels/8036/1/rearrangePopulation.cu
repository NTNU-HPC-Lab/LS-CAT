#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rearrangePopulation(float *gene, float *fit, int* metaData)
{
const int idx = threadIdx.x + blockDim.x*blockIdx.x;
int nGene = metaData[1];
int nHalf = nGene / 2;
if(idx> nHalf) return;

int j = nGene - 1 - idx;

if (fit[idx] < fit[j]) {
for(int k=0; k<6; k++) {
float t = gene[idx*6+k];
gene[idx*6+k] = gene[j*6+k];
gene[j*6+k] = t;
t = fit[idx];
fit[idx] = fit[j];
fit[j] = t;
}
}
}