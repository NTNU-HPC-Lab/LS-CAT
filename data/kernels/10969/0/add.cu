#include "hip/hip_runtime.h"
#include "includes.h"

// Define and implement the GPU addition function
// This version is a vector addition, with N threads
// and and N blocks
// Adding one a and b instance and storing in one c instance.

// Nmber of blocks
#define N (2048*2048)
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c)
{
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}