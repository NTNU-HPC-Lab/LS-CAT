#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NodesApplyTramplingEffectKernel(float* target, float* distanceToPath, int graphW, int graphH, float pathThickness, float tramplingCoefficient)
{
int i = 1 + blockIdx.x * blockDim.x + threadIdx.x;
int j = 1 + blockIdx.y * blockDim.y + threadIdx.y;
if(i <= graphW && j <= graphH) {
int index = i + j * (graphW + 2);

float t = distanceToPath[index];
t = max(0.0f, min(1.0f, fabsf(t / pathThickness)));
t = t * (t * (-4 * t + 6) - 3) + 1;		// cubic parabola

atomicAdd(&target[index], t * tramplingCoefficient);
}
}