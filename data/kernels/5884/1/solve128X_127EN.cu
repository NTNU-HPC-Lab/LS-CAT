#include "hip/hip_runtime.h"
#include "includes.h"
/// System includes

// CUDA runtime

#define threadsPerBlock  (512)
#define MaxCuckooNum (4*4096)
#define MaxGpuNum (1024)
#define trim (32)
#define SolveThreadsPerBlock (128)
#define SolveEN (128)
#define CuckooNum (2*4096)

#define rotl(x, b) (((x) << (b)) | ((x) >> (64 - (b))))
#define EBIT 15
#define CLEN 12
#define EN (1 << EBIT)
#define M (EN << 1)
#define MASK ((1 << EBIT) - 1)
#define CN CLEN << 2

struct GPU_DEVICE
{
uint32_t cproof[CuckooNum][CLEN];
uint8_t msg[CuckooNum][32];
uint8_t alive[CuckooNum][EN >> 3];
uint8_t calive[CuckooNum][EN >> 3];
uint64_t nonces[CuckooNum];

uint8_t  *gmsg = NULL;
uint8_t  *gRHash = NULL;
uint32_t *gRege = NULL;
uint32_t *gproof = NULL;
uint32_t *gnode = NULL;
};

GPU_DEVICE *gpu_divices[MaxGpuNum] = {NULL};
uint32_t gpu_divices_cnt = 0;

// set siphash keys from 32 byte char array
#define setkeys() \
k0 = (((uint64_t *)mesg)[0]); \
k1 = (((uint64_t *)mesg)[1]); \
k2 = (((uint64_t *)mesg)[2]); \
k3 = (((uint64_t *)mesg)[3]);

#define sip_round() \
v0 += v1; v2 += v3; v1 = rotl(v1, 13); \
v3 = rotl(v3, 16); v1 ^= v0; v3 ^= v2; \
v0 = rotl(v0, 32); v2 += v1; v0 += v3; \
v1 = rotl(v1, 17); v3 = rotl(v3, 21); \
v1 ^= v2; v3 ^= v0; v2 = rotl(v2, 32);

__global__ void solve128X_127EN(uint32_t *gRege, uint8_t *gRHash, uint32_t *gproof)
{
unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
uint32_t i, tmp;
uint8_t u, v;

uint32_t block_tid = id % SolveThreadsPerBlock;
uint32_t *Rege = gRege + id * SolveEN;
uint8_t *RHash = gRHash + id * (SolveEN << 1);
uint32_t *proof = gproof + id * CLEN;

__shared__ uint32_t path[SolveThreadsPerBlock][CLEN];
__shared__ uint8_t graph[SolveThreadsPerBlock][SolveEN << 1];

uint8_t pre;
uint8_t cur;
uint8_t next;

memset(&graph[block_tid], 0xff, (SolveEN << 1));
proof[0] = 0xffffffff;

for (i = 0; i<SolveEN; i++)
{
if (Rege[i] == 0xffffffff)
{
break;
}
u = RHash[i << 1];
v = RHash[(i << 1) + 1];
__syncthreads();
pre = 0xff;
cur = u;
while (cur != 0xff)
{
next = graph[block_tid][cur];
graph[block_tid][cur] = pre;
pre = cur;
cur = next;
}
int m = 0;
cur = v;
while (graph[block_tid][cur] != 0xff && m < CLEN)
{
cur = graph[block_tid][cur];
++m;
}
if (cur != u)
{
graph[block_tid][u] = v;
}
else if (m == CLEN - 1)
{
int j;
cur = v;
for (j = 0; j <= m; ++j)
{
path[block_tid][j] = cur;
cur = graph[block_tid][cur];
}

memset(&graph[block_tid], 0xff, (SolveEN << 1));

for (j = 1; j <= m; ++j)
{
graph[block_tid][path[block_tid][j]] = path[block_tid][j - 1];
}

int k = 0;
int b = CLEN - 1;
for (j = 0; k < b; ++j)
{
u = RHash[j << 1];
v = RHash[(j << 1) + 1];
if (graph[block_tid][u] == v)
{
path[block_tid][k] = Rege[j];
graph[block_tid][u] = 0xff;
++k;
}
else if(graph[block_tid][v] == u)
{
path[block_tid][k] = Rege[j];
graph[block_tid][v] = 0xff;
++k;
}
}
path[block_tid][k] = Rege[i];

for (j = 0; j < CLEN - 1; j++) // sort
{
for (k = 0; k < CLEN - j - 1; k++)
{
if (path[block_tid][k]>path[block_tid][k + 1])
{
tmp = path[block_tid][k];
path[block_tid][k] = path[block_tid][k + 1];
path[block_tid][k + 1] = tmp;
}
}
}
for (j = 0; j < CLEN; j++)proof[j] = path[block_tid][j];
break;
}
}
__syncthreads();
}