#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void hoCalc(double* rn, double* soilHeat, double* ho, int width_band) {

int col = threadIdx.x + blockIdx.x * blockDim.x;

while (col < width_band) {

ho[col] = rn[col] - soilHeat[col];

col += blockDim.x * gridDim.x;

}

}