#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
return;

long int id = (k * c_Size.y + j) * c_Size.x + i;

float norm = sqrt(grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id]);
if (norm > gamma)
{
float ratio = gamma / norm;
grad_x[id] *= ratio;
grad_y[id] *= ratio;
grad_z[id] *= ratio;
}
}