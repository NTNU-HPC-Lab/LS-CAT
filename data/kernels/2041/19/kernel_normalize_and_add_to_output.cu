#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
{
return;
}

// Index row major into the volume
long int out_idx = i + (j + k * c_volSize.y) * (c_volSize.x);

float eps = 1e-6;

// Divide the output volume's voxels by the accumulated splat weights
//   unless the accumulated splat weights are equal to zero
if (c_normalize)
{
if (abs(dev_accumulate_weights[out_idx]) > eps)
dev_vol_out[out_idx] = dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / dev_accumulate_weights[out_idx]);
else
dev_vol_out[out_idx] = dev_vol_in[out_idx];
}
else
dev_vol_out[out_idx] = dev_vol_in[out_idx] + dev_accumulate_values[out_idx];
}