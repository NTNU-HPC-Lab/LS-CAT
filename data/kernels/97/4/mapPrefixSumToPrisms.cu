#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapPrefixSumToPrisms( const unsigned numberOfPrisms, const unsigned raysPerSample, const unsigned reflectionSlices, const unsigned* raysPerPrism, const unsigned* prefixSum, unsigned *indicesOfPrisms, unsigned *numberOfReflections ){

int id = threadIdx.x + (blockIdx.x * blockDim.x);
// break if we have too many threads (this is likely)
if(id >= numberOfPrisms*reflectionSlices) return;

const unsigned count            = raysPerPrism[id];
const unsigned startingPosition = prefixSum[id];
const unsigned reflection_i     = id / numberOfPrisms;
const unsigned prism_i          = id % numberOfPrisms;

for(unsigned i=0; i < count ; ++i){
indicesOfPrisms[startingPosition + i] = prism_i;
numberOfReflections[startingPosition + i] = reflection_i;
}
}