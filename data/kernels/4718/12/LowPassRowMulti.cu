#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
__shared__ float data[CONVROW_W + 2*RADIUS];
const int tx = threadIdx.x;
const int block = blockIdx.x/(NUM_SCALES+3);
const int scale = blockIdx.x - (NUM_SCALES+3)*block;
const int xout = block*CONVROW_W + tx;
const int loadPos = xout - RADIUS;
const int yptr = blockIdx.y*pitch;
const int writePos = yptr + height*pitch*scale + xout;
float *kernel = d_Kernel + scale*16;

if (loadPos<0)
data[tx] = d_Data[yptr];
else if (loadPos>=width)
data[tx] = d_Data[yptr + width-1];
else
data[tx] = d_Data[yptr + loadPos];
__syncthreads();
if (xout<width && tx<CONVROW_W)
d_Result[writePos] =
(data[tx+0] + data[tx+8])*kernel[0] +
(data[tx+1] + data[tx+7])*kernel[1] +
(data[tx+2] + data[tx+6])*kernel[2] +
(data[tx+3] + data[tx+5])*kernel[3] +
data[tx+4]*kernel[4];
__syncthreads();
}