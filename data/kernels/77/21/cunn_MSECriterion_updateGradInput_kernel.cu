#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
int k = blockIdx.x;
float *gradInput_k = gradInput + k*dim;
float *input_k = input + k*dim;
float *target_k = target + k*dim;

int i_start = threadIdx.x;
int i_end = dim;
int i_step = blockDim.x;

// gradInput
for (int i=i_start; i<i_end; i+=i_step)
gradInput_k[i] = norm*(input_k[i] - target_k[i]);
}