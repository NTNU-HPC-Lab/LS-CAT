#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
// iterators
int xx, yy;

// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;
//int k = blockIdx.x % input_n;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
gradOutput = gradOutput + o*output_w*output_h;
gradInput = gradInput + i*input_w*input_h;
indices_x = indices_x + o*output_w*output_h;
indices_y = indices_y + o*output_w*output_h;

// compute gradInput
for(yy = yy_start; yy < yy_end; yy+=yy_step) {
for(xx = xx_start; xx < xx_end; xx+=xx_step) {
float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
float *ptr_gradOutput = gradOutput + yy*output_w + xx;
float *ptr_ind_x = indices_x + yy*output_w + xx;
float *ptr_ind_y = indices_y + yy*output_w + xx;
float z = *ptr_gradOutput;

int argmax_x = (*ptr_ind_x)-1;
int argmax_y = (*ptr_ind_y)-1;

ptr_gradInput[argmax_x + argmax_y*input_w] += z;
}
}
}