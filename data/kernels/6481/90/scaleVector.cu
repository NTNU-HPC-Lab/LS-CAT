#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
const int pos = blockIdx.x * blockDim.x + threadIdx.x;

if (pos >= len) return;

d_res[pos] = d_src[pos] * scale;
}