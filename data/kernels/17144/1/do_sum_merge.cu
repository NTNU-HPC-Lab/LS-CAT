#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void do_sum_merge(int *datas, int n){
int tid=blockDim.x*threadIdx.y+threadIdx.x;
//int idx=blockIdx.x*blockDim.x+threadIdx.x;
//int idy=blockIdx.y*blockDim.y+threadIdx.y;
//int bid=gridDim.x*blockDim.x*idy+idx;
while(n>1){
if (tid< (1+n)/2 && n-1-tid!=tid){
datas[tid]+=datas[n-1-tid];
printf ("%d->%d->%d\n",n,tid,datas[tid]);
}
n/=2;
__syncthreads();
}
}