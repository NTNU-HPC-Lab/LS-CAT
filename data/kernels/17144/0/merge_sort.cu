#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

// 用宏变长参数来实现
__global__ void merge_sort(int *datas, int n){
int tid=blockDim.x*threadIdx.y+threadIdx.x;
extern __shared__ int shared[];
if (tid<n) shared[tid] = datas[tid];
__syncthreads();
int cnt=1;
for (int gap=2; gap<n*2; gap<<=1, cnt++){
if (tid%gap==0){
int left=tid+n*((cnt+1)%2);
int mid=tid+gap/2+n*((cnt+1)%2);
int right=mid;
int end=tid+gap+((cnt+1)%2)*n;
int full_end=(1+(cnt+1)%2)*n;
int res_ind=n*(cnt%2)+tid;

while((left<mid && left<full_end) || (right<end && right<full_end)){
if (!(left<mid && left<full_end)){
shared[res_ind]=shared[right];
right++;
}else if (!(right<end && right<full_end)){
shared[res_ind]=shared[left];
left++;
}else{
if (shared[right]> shared[left]){
shared[res_ind]=shared[left];
left++;
}else{
shared[res_ind]=shared[right];
right++;
}
}
res_ind++;
}
}
__syncthreads();
}

datas[tid]=shared[tid+ ((cnt+1)%2)*n];
}