#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matriz_GPU_Mult(int *a, int *b, int *c) {
int k, sum = 0;
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < N && j < N) {
for (k = 0; k < N; k++) {
sum += a[j * N + k] * b[k * N + i];
}
c[j * N + i] = sum;
}
}