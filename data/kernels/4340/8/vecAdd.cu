#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, int offset, int len) {
//@@ Insert code to implement vector addition here
int i =  threadIdx.x;

if( (offset + i) <len ) in1[offset + i] = in1[offset + i]+in1[offset-1];
if( (offset + i + blockDim.x ) <len ) in1[offset + i+ blockDim.x] = in1[offset + i+ blockDim.x]+in1[offset-1];
}