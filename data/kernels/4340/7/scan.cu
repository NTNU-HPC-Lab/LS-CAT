#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float * input, float * output,  int len) {
//@@ Load a segment of the input vector into shared memory
__shared__ float sh_input[2048];


int tx = threadIdx.x;
int tx2= tx + blockDim.x;
int bdimx = blockDim.x;
int i = 2*blockIdx.x*blockDim.x + tx;

int start = 2*blockIdx.x*blockDim.x;
int Col1 = start + tx;
int Col2 = start + bdimx + tx;

if( Col2 < len)
{
// Collaborative loading of A
sh_input[tx] = input[ Col1];
sh_input[tx2] = input[ Col2];
}
else if ( Col1 < len)
{	// Control divergence at the edge
sh_input[tx] = input[ Col1];
sh_input[tx2]= 0.0f;
}
else
{	// Control divergence at the edge
sh_input[tx] = 0.0f;
sh_input[tx2]= 0.0f;
}
__syncthreads();
//output[Col1] = sh_input[tx]; output[Col2] = sh_input[tx2];

unsigned int stride; int index;
//  @@ Traverse the reduction tree down
for (stride = 1;stride <= 2*bdimx ; stride *= 2)
{
index = (tx +1)* stride*2 -1;
if (index  < 2*bdimx)
sh_input[index] += sh_input[index-stride];
__syncthreads();
}

//@@ Traverse the reduction tree up
for ( stride = bdimx/2;  stride > 0; stride/=2)
{
__syncthreads();
index = (tx +1)* stride*2 -1;
if (index + stride < 2*bdimx)
sh_input[index+stride] += sh_input[index];
}

//@@ Write the computed sum of the block to the output vector at the
//@@ correct index
__syncthreads();
output[i] = sh_input[tx];
if ( i + bdimx < len)
{
output[i + bdimx] = sh_input[tx2];
}

}