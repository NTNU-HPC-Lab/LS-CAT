#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
//@@ Insert code to implement matrix multiplication here
int iRow = blockIdx.y*blockDim.y+threadIdx.y;
int iCol = blockIdx.x*blockDim.x+threadIdx.x;
if(( iRow < numARows) && (iCol < numBColumns)) {
float Cvalue = 0.0;
for (int i = 0;i< numAColumns;++i)
{
Cvalue += A[iRow*numAColumns+i]*B[iCol+i*numBColumns];
}
C[iRow*numBColumns+iCol] = Cvalue;
}
}