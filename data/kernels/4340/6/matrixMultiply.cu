#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
//@@ Insert code to implement matrix multiplication here
__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int Row = by *TILE_WIDTH  + ty;
int Col = bx * TILE_WIDTH + tx;
//int Row = blockIdx.y*blockDim.y+threadIdx.y;
//int Col = blockIdx.x*blockDim.x+threadIdx.x;
float Cvalue = 0;

// Loop over the A and B tiles required to compute the C element
for (int t = 0; t < (numBRows-1)/TILE_WIDTH + 1; ++t)
{
if(Row < numARows && t*TILE_WIDTH+tx < numBRows)
{
// Collaborative loading of A
ds_A[ty][tx] = A[Row*numAColumns + t*TILE_WIDTH+tx];
}
else
{	// Control divergence at the edge
ds_A[ty][tx]= 0.0;
}

if ( t*TILE_WIDTH+ty < numBRows && Col < numBColumns)
{
// Collaborative loading of B if within range of matrix
ds_B[ty][tx] = B[(t*TILE_WIDTH+ty)*numBColumns + Col];
}
else
{
ds_B[ty][tx] = 0.0;
}

__syncthreads();

for (int i = 0; i < TILE_WIDTH; ++i)
{
Cvalue += ds_A[ty][i] * ds_B[i][tx];
}
__syncthreads();
}
if ( Row < numARows && Col < numBColumns)
C[Row*numBColumns+Col] = Cvalue;


}