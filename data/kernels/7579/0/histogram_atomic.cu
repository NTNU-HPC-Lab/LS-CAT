#include "hip/hip_runtime.h"
#include "includes.h"


#define DATA_SIZE (1024 * 1024 * 256)
#define DATA_RANGE (256)

void printHist(int * arr, char * str);




__global__ void histogram_atomic(float * a, int * histo, int n) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid >= n) return;
atomicAdd(histo + (int)a[tid], 1);
}