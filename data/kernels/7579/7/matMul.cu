#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(float *a, float *b, float *c, int M, int N, int K)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if(row >= M || col >= N)
return;

float sum = 0.f;

__syncthreads();

for(int k = 0; k < K; k++)
{
sum += a[col * K + k] * b[k * N + row];
}

__syncthreads();

c[col * N + row] = sum;
}