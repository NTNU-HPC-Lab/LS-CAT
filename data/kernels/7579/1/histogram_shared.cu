#include "hip/hip_runtime.h"
#include "includes.h"


#define DATA_SIZE (1024 * 1024 * 256)
#define DATA_RANGE (256)

void printHist(int * arr, char * str);




__global__ void histogram_shared(float * a, int * histo, int n) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
__shared__ int sh[DATA_RANGE];

if(threadIdx.x < 256) sh[threadIdx.x] = 0;
__syncthreads();

if(tid < n) atomicAdd(&sh[(int)a[tid]], 1);
__syncthreads();

if(threadIdx.x < 256) atomicAdd(&histo[threadIdx.x], sh[threadIdx.x]);

}