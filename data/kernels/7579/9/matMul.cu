#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(double *a, double *b, double *c)
{
const int NUM_THREAD_IN_BLOCK = blockDim.x * blockDim.y * blockDim.z;

int bID = blockIdx.z * (gridDim.y * gridDim.x * NUM_THREAD_IN_BLOCK) + blockIdx.y * (gridDim.x * NUM_THREAD_IN_BLOCK) + (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z));
int tID = bID + ((blockDim.y * blockDim.x) * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;


for(int i = 0; i < MATRIX_J; i++)
c[tID] += a[(tID * MATRIX_J) + i] * b[i];
}