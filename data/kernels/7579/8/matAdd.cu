#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matAdd(int *a, int *b, int *c)
{
const int NUM_THREAD_IN_BLOCK = blockDim.x * blockDim.y * blockDim.z;

int bID = blockIdx.z * (gridDim.y * gridDim.x * NUM_THREAD_IN_BLOCK) + blockIdx.y * (gridDim.x * NUM_THREAD_IN_BLOCK) + (blockIdx.x * NUM_THREAD_IN_BLOCK);
int tID = bID + ((blockDim.y * blockDim.x) * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

c[tID] = a[tID] + b[tID];
}