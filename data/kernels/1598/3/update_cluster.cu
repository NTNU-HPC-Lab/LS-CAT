#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_cluster(int *cluster, float *centroid, float *B_c, float *G_c, float *R_c, int size_image, int n_threads, int k,  float *Bdata, float *Gdata, float *Rdata, float *nValue)
{

unsigned int tid = threadIdx.x;

int size_per_thread = int(size_image/n_threads);
int start = tid*size_per_thread;
int end = start + size_per_thread;

float count = 0;
float B = 0;
float G = 0;
float R = 0;

if (tid >=size_image){ return; }

if (tid==n_threads-1)
{
start = (n_threads-1)*size_per_thread;
end = size_image;
}
for(int j = start; j < end; j++)
{
if(cluster[j] == k)
{
B = B + (B_c[j]);
G = G + (G_c[j]);
R = R + (R_c[j]);
count = count + 1;
}
}

nValue[tid] = count;
Bdata[tid] = B;
Gdata[tid] = G;
Rdata[tid] = R;

__syncthreads();

for(unsigned int s=1; s < blockDim.x; s *= 2)
{
if(tid % (2*s) == 0)
{
nValue[tid] += nValue[tid + s];
Bdata[tid] += Bdata[tid + s];
Gdata[tid] += Gdata[tid + s];
Rdata[tid] += Rdata[tid + s];
}
__syncthreads();
}

if(tid == 0)
{
if (nValue[0] != 0)
{
centroid[k*3 + 0] = Bdata[0] / nValue[0];
centroid[k*3 + 1] = Gdata[0] / nValue[0];
centroid[k*3 + 2] = Rdata[0] / nValue[0];
}
}
}