#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_cluster(int *cluster, float *centroid, float *B_c, float *G_c, float *R_c, int size_image, int n_threads, int K, float *Bdata, float *Gdata, float *Rdata, float *nValue)
{

unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int id = threadIdx.x;
int k = blockIdx.x;

int size_per_thread = int(size_image/n_threads);
int start = id*size_per_thread;
int end = start + size_per_thread;

float count = 0;
float B = 0;
float G = 0;
float R = 0;

nValue[tid] = 0;
Bdata[tid] = 0;
Gdata[tid] = 0;
Rdata[tid] = 0;

if (id >=size_image){ return; }

if (id==n_threads-1)
{
start = (n_threads-1)*size_per_thread;
end = size_image;
}
for(int j = start; j < end; j++)
{
if(cluster[j] == k)
{
B = B + (B_c[j]);
G = G + (G_c[j]);
R = R + (R_c[j]);
count = count + 1;
}
}

nValue[tid] = count;
Bdata[tid] = B;
Gdata[tid] = G;
Rdata[tid] = R;

__syncthreads();

for(unsigned int s=1; s < blockDim.x; s *= 2)
{
if(tid % (2*s) == 0 && tid+s<=(k+1)*n_threads)
{
nValue[tid] += nValue[tid + s];
Bdata[tid] += Bdata[tid + s];
Gdata[tid] += Gdata[tid + s];
Rdata[tid] += Rdata[tid + s];
}
__syncthreads();
}


if(id == 0)
{
//printf("Block: %d. nValue: %f.\n", k, nValue[tid]);
if (nValue[tid] != 0)
{
centroid[k*3 + 0] = Bdata[tid] / nValue[tid];
centroid[k*3 + 1] = Gdata[tid] / nValue[tid];
centroid[k*3 + 2] = Rdata[tid] / nValue[tid];
}
}
}