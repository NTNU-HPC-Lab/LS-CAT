#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel (float *output, float *input, int feature_size, int spatial_size, int data_size, float constant)
{
int idx = (threadIdx.x + blockDim.x*blockIdx.x);
idx = (idx/spatial_size)*feature_size + idx % spatial_size;

if (idx < data_size) {
int next_idx = idx + feature_size;
float logsum = 0.0;
float max = -2e38;
// max
for(int i = idx; i < next_idx; i += spatial_size) {
if (max < input[i]) max = input[i];
}

// logsum
for(int i = idx; i < next_idx; i += spatial_size) {
if (!isnan(input[i])) {
logsum += __expf(input[i]-max);
}
}
logsum += constant;
logsum = __logf(logsum) + max;

// logsoftmax
for(int i = idx; i < next_idx; i += spatial_size){
output[i] = input[i] - logsum;
}
}
}