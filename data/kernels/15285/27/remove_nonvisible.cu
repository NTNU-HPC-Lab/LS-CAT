#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void remove_nonvisible(float *y, int size, int size3)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int x = id % size3;
if (y[id] >= x) {
y[id] = 0;
}
}
}