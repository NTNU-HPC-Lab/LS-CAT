#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 128

#define DISP_MAX 256

__global__ void remove_white(float *x, float *y, int size)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
if (x[id] == 255) {
y[id] = 0;
}
}
}