#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecDiv(float* a,float* b,float* c,const int N){
//const int i = blockIdx.x*blockDim.x + threadIdx.x;
const int i = gridDim.x*blockDim.x*blockIdx.y +  blockIdx.x*blockDim.x + threadIdx.x;
if(i<N)
c[i] = a[i]/b[i];
//c[i] = __fdividef(a[i],b[i]);  //faster, less-accurate divide
}