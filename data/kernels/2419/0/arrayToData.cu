#include "hip/hip_runtime.h"
#include "includes.h"



//no performance difference if using float Mono input instead of float4 RGBA
//texture<float, hipTextureType2D, hipReadModeElementType> inTex;
//g_odata[offset] = tex2D(inTex, xc, yc);

texture<float4, hipTextureType2D, hipReadModeElementType> inTex;
surface<void, cudaSurfaceType2D> outputSurface;

__global__ void arrayToData(float *g_odata, uint* keys, int imgw, int imgh)
{

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int offset = x + y * imgw;

if (x < imgw && y < imgh) {

float xc = x + 0.5;
float yc = y + 0.5;


g_odata[offset] = tex2D(inTex, xc, yc).x;
keys[offset] = offset;
}


}