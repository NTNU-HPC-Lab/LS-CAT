#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int* a,int* b, int*c )
{

int tid = blockIdx.x * blockDim.x + threadIdx.x;

c[tid] = a[tid] + b[tid];

}