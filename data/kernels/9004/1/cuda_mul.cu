#include "hip/hip_runtime.h"
#include "includes.h"

#define MAT_TYPE double
#define MAT_SIZE 1024
#define N MAT_SIZE
#define N2 MAT_SIZE*MAT_SIZE

#define BLOCK 256
#define THREAD 512

void stopwatch(int);






__global__ void cuda_mul(MAT_TYPE* A,MAT_TYPE* B,MAT_TYPE* C,int w)
{
int tid,tx,ty;

tx = blockDim.x * blockIdx.x + threadIdx.x;
ty = blockDim.y * blockIdx.y + threadIdx.y;
tid = w*ty + tx;

MAT_TYPE v = 0;
MAT_TYPE a = 0;
MAT_TYPE b = 0;

for(int i=0;i< w;i++)
{
a = A[ty * w + i];
b = B[i * w + tx];
v += a+b;
}

C[tid]= v;
}