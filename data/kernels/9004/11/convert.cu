#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convert(double* A,double* C)
{
int idx = BLOCK*blockIdx.x + threadIdx.x;
int i;
int stride = BLOCK * THREAD;

for(i=idx;i<SIZE;i+=stride)
A[i] = C[SIZE-i-1];

}