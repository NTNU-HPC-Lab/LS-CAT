#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_mul(int* A, int* B, int* C, int w)
{
int tid,tx,ty;

//range of tx,ty 0 ~ w
tx = blockDim.x * blockIdx.x + threadIdx.x;
ty = blockDim.y * blockIdx.y + threadIdx.y;
tid = w*ty + tx;

int v = 0;
int a = 0;
int b = 0;


/*
oooo    oxo
xxxx  X oxo
oooo    oxo
oxo
*/

for(int i=0;i< w;i++)
{
a = A[ty * w + i];
b = B[i * w + tx];
v += a+b;
}

C[tid]= v;
}