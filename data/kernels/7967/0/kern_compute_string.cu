#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// includes

extern "C"
{
}
#define MEMSIZE 30


/* Function computing the final string to print */
__global__ void kern_compute_string(char *res, char *a, char *b, char *c, int length)
{
int i;
i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < length)
{
res[i] = a[i] + b[i] + c[i];
}
}