#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 1e-4

__global__ void bilateral_smooth_kernel( float *affine_model, float *filtered_affine_model, float *guide, int h, int w, int kernel_radius, float sigma1, float sigma2 )
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
if (id < size) {
int x = id % w;
int y = id / w;

double sum_affine[12] = {};
double sum_weight = 0;
for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
int yy = y + dy, xx = x + dx;
int id2 = yy * w + xx;
if (0 <= xx && xx < w && 0 <= yy && yy < h) {
float color_diff1 = guide[yy*w + xx] - guide[y*w + x];
float color_diff2 = guide[yy*w + xx + size] - guide[y*w + x + size];
float color_diff3 = guide[yy*w + xx + 2*size] - guide[y*w + x + 2*size];
float color_diff_sqr =
(color_diff1*color_diff1 + color_diff2*color_diff2 + color_diff3*color_diff3) / 3;

float v1 = exp(-(dx * dx + dy * dy) / (2 * sigma1 * sigma1));
float v2 = exp(-(color_diff_sqr) / (2 * sigma2 * sigma2));
float weight = v1 * v2;

for (int i = 0; i < 3; i++) {
for (int j = 0; j < 4; j++) {
int affine_id = i * 4 + j;
sum_affine[affine_id] += weight * affine_model[id2*12 + affine_id];
}
}
sum_weight += weight;
}
}
}

for (int i = 0; i < 3; i++) {
for (int j = 0; j < 4; j++) {
int affine_id = i * 4 + j;
filtered_affine_model[id*12 + affine_id] = sum_affine[affine_id] / sum_weight;
}
}
}
return ;
}