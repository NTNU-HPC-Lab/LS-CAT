#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 1e-4

__device__ bool InverseMat4x4(double m_in[4][4], double inv_out[4][4]) {
double m[16], inv[16];
for (int i = 0; i < 4; i++) {
for (int j = 0; j < 4; j++) {
m[i * 4 + j] = m_in[i][j];
}
}

inv[0] = m[5]  * m[10] * m[15] -
m[5]  * m[11] * m[14] -
m[9]  * m[6]  * m[15] +
m[9]  * m[7]  * m[14] +
m[13] * m[6]  * m[11] -
m[13] * m[7]  * m[10];

inv[4] = -m[4]  * m[10] * m[15] +
m[4]  * m[11] * m[14] +
m[8]  * m[6]  * m[15] -
m[8]  * m[7]  * m[14] -
m[12] * m[6]  * m[11] +
m[12] * m[7]  * m[10];

inv[8] = m[4]  * m[9] * m[15] -
m[4]  * m[11] * m[13] -
m[8]  * m[5] * m[15] +
m[8]  * m[7] * m[13] +
m[12] * m[5] * m[11] -
m[12] * m[7] * m[9];

inv[12] = -m[4]  * m[9] * m[14] +
m[4]  * m[10] * m[13] +
m[8]  * m[5] * m[14] -
m[8]  * m[6] * m[13] -
m[12] * m[5] * m[10] +
m[12] * m[6] * m[9];

inv[1] = -m[1]  * m[10] * m[15] +
m[1]  * m[11] * m[14] +
m[9]  * m[2] * m[15] -
m[9]  * m[3] * m[14] -
m[13] * m[2] * m[11] +
m[13] * m[3] * m[10];

inv[5] = m[0]  * m[10] * m[15] -
m[0]  * m[11] * m[14] -
m[8]  * m[2] * m[15] +
m[8]  * m[3] * m[14] +
m[12] * m[2] * m[11] -
m[12] * m[3] * m[10];

inv[9] = -m[0]  * m[9] * m[15] +
m[0]  * m[11] * m[13] +
m[8]  * m[1] * m[15] -
m[8]  * m[3] * m[13] -
m[12] * m[1] * m[11] +
m[12] * m[3] * m[9];

inv[13] = m[0]  * m[9] * m[14] -
m[0]  * m[10] * m[13] -
m[8]  * m[1] * m[14] +
m[8]  * m[2] * m[13] +
m[12] * m[1] * m[10] -
m[12] * m[2] * m[9];

inv[2] = m[1]  * m[6] * m[15] -
m[1]  * m[7] * m[14] -
m[5]  * m[2] * m[15] +
m[5]  * m[3] * m[14] +
m[13] * m[2] * m[7] -
m[13] * m[3] * m[6];

inv[6] = -m[0]  * m[6] * m[15] +
m[0]  * m[7] * m[14] +
m[4]  * m[2] * m[15] -
m[4]  * m[3] * m[14] -
m[12] * m[2] * m[7] +
m[12] * m[3] * m[6];

inv[10] = m[0]  * m[5] * m[15] -
m[0]  * m[7] * m[13] -
m[4]  * m[1] * m[15] +
m[4]  * m[3] * m[13] +
m[12] * m[1] * m[7] -
m[12] * m[3] * m[5];

inv[14] = -m[0]  * m[5] * m[14] +
m[0]  * m[6] * m[13] +
m[4]  * m[1] * m[14] -
m[4]  * m[2] * m[13] -
m[12] * m[1] * m[6] +
m[12] * m[2] * m[5];

inv[3] = -m[1] * m[6] * m[11] +
m[1] * m[7] * m[10] +
m[5] * m[2] * m[11] -
m[5] * m[3] * m[10] -
m[9] * m[2] * m[7] +
m[9] * m[3] * m[6];

inv[7] = m[0] * m[6] * m[11] -
m[0] * m[7] * m[10] -
m[4] * m[2] * m[11] +
m[4] * m[3] * m[10] +
m[8] * m[2] * m[7] -
m[8] * m[3] * m[6];

inv[11] = -m[0] * m[5] * m[11] +
m[0] * m[7] * m[9] +
m[4] * m[1] * m[11] -
m[4] * m[3] * m[9] -
m[8] * m[1] * m[7] +
m[8] * m[3] * m[5];

inv[15] = m[0] * m[5] * m[10] -
m[0] * m[6] * m[9] -
m[4] * m[1] * m[10] +
m[4] * m[2] * m[9] +
m[8] * m[1] * m[6] -
m[8] * m[2] * m[5];

double det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

if (abs(det) < 1e-9) {
return false;
}


det = 1.0 / det;

for (int i = 0; i < 4; i++) {
for (int j = 0; j < 4; j++) {
inv_out[i][j] = inv[i * 4 + j] * det;
}
}

return true;
}
__global__ void best_local_affine_kernel( float *output, float *input, float *affine_model, int h, int w, float epsilon, int kernel_radius )
{
int size = h * w;
int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < size) {
int x = id % w, y = id / w;

double Mt_M[4][4] = {}; // 4x4
double invMt_M[4][4] = {};
double Mt_S[3][4] = {}; // RGB -> 1x4
double A[3][4] = {};
for (int i = 0; i < 4; i++)
for (int j = 0; j < 4; j++) {
Mt_M[i][j] = 0, invMt_M[i][j] = 0;
if (i != 3) {
Mt_S[i][j] = 0, A[i][j] = 0;
if (i == j)
Mt_M[i][j] = 1e-3;
}
}

for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {

int xx = x + dx, yy = y + dy;
int id2 = yy * w + xx;

if (0 <= xx && xx < w && 0 <= yy && yy < h) {

Mt_M[0][0] += input[id2 + 2*size] * input[id2 + 2*size];
Mt_M[0][1] += input[id2 + 2*size] * input[id2 + size];
Mt_M[0][2] += input[id2 + 2*size] * input[id2];
Mt_M[0][3] += input[id2 + 2*size];

Mt_M[1][0] += input[id2 + size] * input[id2 + 2*size];
Mt_M[1][1] += input[id2 + size] * input[id2 + size];
Mt_M[1][2] += input[id2 + size] * input[id2];
Mt_M[1][3] += input[id2 + size];

Mt_M[2][0] += input[id2] * input[id2 + 2*size];
Mt_M[2][1] += input[id2] * input[id2 + size];
Mt_M[2][2] += input[id2] * input[id2];
Mt_M[2][3] += input[id2];

Mt_M[3][0] += input[id2 + 2*size];
Mt_M[3][1] += input[id2 + size];
Mt_M[3][2] += input[id2];
Mt_M[3][3] += 1;

Mt_S[0][0] += input[id2 + 2*size] * output[id2 + 2*size];
Mt_S[0][1] += input[id2 + size] * output[id2 + 2*size];
Mt_S[0][2] += input[id2] * output[id2 + 2*size];
Mt_S[0][3] += output[id2 + 2*size];

Mt_S[1][0] += input[id2 + 2*size] * output[id2 + size];
Mt_S[1][1] += input[id2 + size] * output[id2 + size];
Mt_S[1][2] += input[id2] * output[id2 + size];
Mt_S[1][3] += output[id2 + size];

Mt_S[2][0] += input[id2 + 2*size] * output[id2];
Mt_S[2][1] += input[id2 + size] * output[id2];
Mt_S[2][2] += input[id2] * output[id2];
Mt_S[2][3] += output[id2];
}
}
}

bool success = InverseMat4x4(Mt_M, invMt_M);

for (int i = 0; i < 3; i++) {
for (int j = 0; j < 4; j++) {
for (int k = 0; k < 4; k++) {
A[i][j] += invMt_M[j][k] * Mt_S[i][k];
}
}
}

for (int i = 0; i < 3; i++) {
for (int j = 0; j < 4; j++) {
int affine_id = i * 4 + j;
affine_model[12 * id + affine_id] = A[i][j];
}
}



}
return ;
}