#include "hip/hip_runtime.h"
#include "includes.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if ((i >= max_x) || (j >= max_y)) return;

int m = (i + j) / 100;
if (m > 0) return;
int k = (i + j) % 100;

//if (i >= max_x) return;
//int pixel_index = j * max_x + i;
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1995, k, 0, &rand_state[k]);
}