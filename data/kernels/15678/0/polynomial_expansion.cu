#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void polynomial_expansion (float* poly, int degree, int n, float* array) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if( index < n )
{
float out = 0.0;
float xtothepowerof = 1.0;
for ( int x = 0; x <= degree; ++x)
{
out += xtothepowerof * poly[x];
xtothepowerof *= array[index];
}
array[index] = out;
}
}