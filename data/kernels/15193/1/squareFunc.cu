#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squareFunc(unsigned int *d_in, unsigned int *d_out)
{
int idx = threadIdx.x;
unsigned int val = d_in[idx];
d_out[idx] = val * val;
//printf("%d square value %d \n  ", idx, d_out[idx]);
}