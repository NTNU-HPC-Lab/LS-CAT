#include "hip/hip_runtime.h"
#include "includes.h"

const int NUMTHREADS = 1024;
int startNodeNumber;
int endNodeNumber;

typedef struct lList {
int path[50];
struct lList *next;
} lList;

__global__ void GPUMultiplyMatrix(long *matrix1, long *matrix2, int paths, int count) {
int element = blockIdx.x * blockDim.x + threadIdx.x;
int i;
while (paths > 0) {
long sum = 0;
int col = element % count;
int row = element / count;
for (i = 0; i < count; i++) {
sum += matrix1[count * i + col] * matrix2[row * count + i];
}
//Wait till all GPU cores are finished
__syncthreads();
matrix2[element] = sum;

paths--;
}
}