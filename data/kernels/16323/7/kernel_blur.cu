#include "hip/hip_runtime.h"
#include "includes.h"
__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
return row * W + col;
}
__global__ void kernel_blur(float* d_I, float* d_Ib, int H, int W) {

size_t row = threadIdx.y + blockDim.y * blockIdx.y;
size_t col = threadIdx.x + blockDim.x * blockIdx.x;
size_t idx = GIDX(row, col, H, W);

if (row >= H - KERN_RADIUS || row <= KERN_RADIUS || col >= W - KERN_RADIUS || col <= KERN_RADIUS) {
return;
}

int count = 0;
for (int i = -KERN_RADIUS; i <= KERN_RADIUS; i++) {
for (int j = -KERN_RADIUS; j <= KERN_RADIUS; j++) {
d_Ib[idx] += d_I[GIDX(row + i, col + j, H, W)] * gaussian_kernel[count];
count++;
}
}

}