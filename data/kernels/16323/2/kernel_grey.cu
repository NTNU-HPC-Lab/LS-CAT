#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_grey( float4* d_Iin, float* d_Iout, int numel ) {

size_t col = threadIdx.x + blockDim.x * blockIdx.x;
if (col >= numel) {
return;
}

float4 pixel = d_Iin[col];

d_Iout[col] = 0.2989f * (pixel.x) + 0.5870f * (pixel.y) + 0.1140f * (pixel.z);
}