#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_fill(float4* d_dx1, float val, int numel) {
size_t col = threadIdx.x + blockIdx.x * blockDim.x;
if (col >= numel) { return; }

d_dx1[col].x = val;
d_dx1[col].y = val;
d_dx1[col].z = val;
d_dx1[col].w = val;
}