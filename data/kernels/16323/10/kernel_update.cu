#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_update( float4* d_positions, float4* d_og_positions, float4* d_velocities, float* d_masses, size_t numel) {

size_t col = threadIdx.x + blockIdx.x * blockDim.x;
if (col >= numel) { return; }

float4 velocity = d_velocities[col];

float mag = sqrtf(velocity.x*velocity.x + velocity.y*velocity.y)*0.03;
float pos = min(mag, 0.50f);
d_positions[col] = make_float4(
d_og_positions[col].x,
d_og_positions[col].y,
pos, 0
);
__syncthreads();
}