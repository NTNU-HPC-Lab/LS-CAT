#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void kernel_update_velocities(float4* d_uv, float4* d_velocities_buffer, int numel) {

size_t col = threadIdx.x + blockIdx.x * blockDim.x;
if (col >= numel) { return; }

d_velocities_buffer[col] = make_float4(
d_uv[col].x,
d_uv[col].y,
0,
0
);
__syncthreads();
}