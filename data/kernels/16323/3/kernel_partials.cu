#include "hip/hip_runtime.h"
#include "includes.h"
__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
return row * W + col;
}
__global__ void kernel_partials( float* d_f1ptr, float* d_f1dx, float* d_f1dy, int H, int W ) {

size_t row = threadIdx.y + blockDim.y * blockIdx.y;
size_t col = threadIdx.x + blockDim.x * blockIdx.x;

size_t idx = GIDX(row, col, H, W);
if (row >= H || row <= 1 || col >= W || col <= 1) {
return;
}

float gray_x1 = d_f1ptr[GIDX(row, col - 1, H, W)];
float gray_x2 = d_f1ptr[GIDX(row, col + 1, H, W)];

float gray_y1 = d_f1ptr[GIDX(row - 1, col, H, W)];
float gray_y2 = d_f1ptr[GIDX(row + 1, col, H, W)];

d_f1dx[idx] = (gray_x2 - gray_x1) / 2.0f;
d_f1dy[idx] = (gray_y2 - gray_y1) / 2.0f;
}