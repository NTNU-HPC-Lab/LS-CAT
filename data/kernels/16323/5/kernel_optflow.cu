#include "hip/hip_runtime.h"
#include "includes.h"
__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
return row * W + col;
}
__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {

const size_t row = threadIdx.y + blockDim.y * blockIdx.y;
const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
const size_t idx = GIDX(row, col, H, W);


if (row >= H - 2 || row <= 2 || col >= W - 2 || col <= 2) {
return;
}
__syncthreads();


float dx2 = 0.0f, dy2 = 0.0f;
float dxdy = 0.0f;
float dxdt = 0.0f, dydt = 0.0f;

for (int i = -2; i <= 2; i++) {
for (int j = -2; j <= 2; j++) {
dx2 += d_dx1[GIDX(row + i, col + j, H, W)] * d_dx1[GIDX(row + i, col + j, H, W)];
dy2 += d_dy1[GIDX(row + i, col + j, H, W)] * d_dy1[GIDX(row + i, col + j, H, W)];

dxdy += d_dx1[GIDX(row + i, col + j, H, W)] * d_dy1[GIDX(row + i, col + j, H, W)];

dxdt += d_dx1[GIDX(row + i, col + j, H, W)] * d_dt[GIDX(row + i, col + j, H, W)];
dydt += d_dy1[GIDX(row + i, col + j, H, W)] * d_dt[GIDX(row + i, col + j, H, W)];
}
}

__syncthreads();
float det = dx2 * dy2 - (dxdy * dxdy);
if (abs(det) <= 1.5e-8) { // 1.5e-5 is based on 1/(255*255)
uv[idx].x = 0.0f;
uv[idx].y = 0.0f;
uv1[idx] = uv[idx];
return;
}

__syncthreads();
float trace = dx2 + dy2;
float delta = sqrtf(trace * trace - 4.0f * det); // delta x2

if (isnan(delta) || trace - delta <= 0.0002) {
uv[idx].x = 0.0f;
uv[idx].y = 0.0f;
uv1[idx] = uv[idx];
return;
}

__syncthreads();
// Calculate flow components

uv[idx].x = (dy2 * -dxdt + dxdy * dydt)/det;
uv[idx].y = (dxdy * dxdt - dx2 * dydt)/ det;
uv1[idx] = uv[idx];

}