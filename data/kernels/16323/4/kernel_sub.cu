#include "hip/hip_runtime.h"
#include "includes.h"
__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
return row * W + col;
}
__global__ void kernel_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

size_t row = threadIdx.y + blockDim.y * blockIdx.y;
size_t col = threadIdx.x + blockDim.x * blockIdx.x;
size_t idx = GIDX(row, col, H, W);

if (row >= H || col >= W) {
return;
}

d_dt[idx] = d_f2ptr[idx] - d_f1ptr[idx];

}