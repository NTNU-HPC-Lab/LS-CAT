#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
size_t col = threadIdx.x + blockIdx.x * blockDim.x;
if (col >= numel) { return; }
uchar3 val = d_Iin[col];

d_Iout[col] = make_float4(
val.x / 255.0f,
val.y / 255.0f,
val.z / 255.0f,
1.0f
);
}