#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_update_models(float4* d_positions, float4* d_modelBuffer, int numel) {

size_t col = threadIdx.x + blockIdx.x * blockDim.x;
if (col >= numel) { return; }

d_modelBuffer[col*4+3] = make_float4(
d_positions[col].x,
d_positions[col].y,
d_positions[col].z,
1
);
__syncthreads();
}