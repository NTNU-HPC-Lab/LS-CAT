#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c

__global__ void encode(char *encodedText, char *decodedText)
{
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;
int startEncoded = id * 101;
int startDecoded = id * 4;
int t,finish=startEncoded+100;
// Make sure we do not go out of bounds
if (id < 15360)
{
for(t=startEncoded;t<finish;t++)
{
if(encodedText[t]==',')
{
decodedText[startDecoded]=encodedText[t+1];
startDecoded++;
}
}
}
}