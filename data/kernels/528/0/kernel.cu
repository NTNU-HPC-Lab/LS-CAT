#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

//Check for edges valid to be part of augmented path

//Update frontier

__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, int* cap_max_mat) {
int row_idx = frontier[blockIdx.x+1];
long offset = N * row_idx;

int col_idx = threadIdx.x;
long offset2 = N * col_idx;
if(adj_mat[offset + col_idx] && (cap_mat[offset + col_idx] < cap_max_mat[offset + col_idx]) && !visited[col_idx]) {
new_frontier[col_idx] = true;
par_mat[offset2 + row_idx] = true;
}

if(adj_mat[offset2 + row_idx] && (cap_mat[offset2 + row_idx] > 0) && !visited[col_idx]) {
new_frontier[col_idx] = true;
par_mat[offset2 + row_idx] = true;
}
}