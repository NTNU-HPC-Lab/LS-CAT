#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(float * d_out, float * d_in)
{
int idx = threadIdx.x;
// threadIdx is a C struct (dim3) with 3 members - .x | .y | .z
float f = d_in[idx];
d_out[idx] = f * f;
}