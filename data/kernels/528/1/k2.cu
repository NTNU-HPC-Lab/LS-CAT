#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

//Check for edges valid to be part of augmented path

//Update frontier

__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier) {
int count = 0;
for(int i=0;i<N;i++) {
if(new_frontier[i]) {
new_frontier[i] = false;
frontier[++count] = i;
visited[i] = true;
}
}
frontier[0] = count;
}