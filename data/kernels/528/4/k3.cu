#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k3(const int N, int* augPath, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, bool* adj_mat, int* cap_max_mat, int* maxflow, bool* augFound) {
augFound[0] = false;

//Find the augmented path
augPath[0] = N - 1;
int i = 1, vertex = N - 1;
while(vertex != 0) {
for(int j = 0; j < N; j++) {
if(par_mat[vertex * N + j]) {
vertex = j;
augPath[i] = vertex;
i++;
break;
}
}
}

//Compute the bottleneck for the augmented path
int bottleneck = -1;
for(int i = 0; i < N; i++) {
if(augPath[i] == 0)
break;
else {
int k = augPath[i];
int j = augPath[i + 1];
int freeCap;
if(adj_mat[j * N + k]) {
freeCap = cap_max_mat[j * N + k] - cap_mat[j * N + k];
} else {
freeCap = cap_mat[k * N + j];
}

if(bottleneck == -1)
bottleneck = freeCap;
else if(freeCap < bottleneck)
bottleneck = freeCap;
}
}
maxflow[0] += bottleneck;

//Update capacities in d_cap_mat
for(int i = 0; i < N; i++) {
if(augPath[i] == 0)
break;
else {
int k = augPath[i];
int j = augPath[i + 1];
if(adj_mat[j * N + k]) {
cap_mat[j * N + k] += bottleneck;
} else {
cap_mat[k * N + j] -= bottleneck;
}
}
}

//Initialize par_mat
for(int i=0;i<N*N;i++)
par_mat[i] = false;

//Initialize visited and frontier
for(int i=0;i<N;i++) visited[i] = false;
for(int i=0;i<N;i++) new_frontier[i] = false;

visited[0] = true;
frontier[0] = 1;
frontier[1] = 0;
}