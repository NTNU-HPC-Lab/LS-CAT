#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier, bool* augFound) {
int count = 0;
for(int i=0;i<N;i++) {
if(new_frontier[i]) {
new_frontier[i] = false;
frontier[++count] = i;
visited[i] = true;
}
}
frontier[0] = count;

//Complete search if sink has been reached
for(int i = 0; i < frontier[0]; i++)
if(frontier[i + 1] == (N - 1))
augFound[0] = true;
}