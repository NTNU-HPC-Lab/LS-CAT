#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

if (ptidx < npoints)
{
short2 loc = loc_[ptidx];

x[ptidx] = loc.x * scale;
y[ptidx] = loc.y * scale;
}
}