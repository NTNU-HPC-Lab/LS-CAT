#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Projection(float2 *__restrict__ newVel, float2 *__restrict__ gradPressure, unsigned int simWidth)
{
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

newVel[y*simWidth+x].x -= gradPressure[y*simWidth+x].x;
newVel[y*simWidth+x].y -= gradPressure[y*simWidth+x].y;
}