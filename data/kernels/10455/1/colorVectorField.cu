#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorVectorField( float3 *colors, float3 *colorMap, float2 *__restrict__ field, dim3 blocks, unsigned int simWidth, unsigned int simHeight)
{
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

int quadIdx = x + simWidth*y;
if (x == simWidth/2 + 10 && y == simWidth/2 + 10)
printf("coloring vector field values on the order of: %f, %f\n", field[quadIdx].x, field[quadIdx].y);

//float mag = sqrt(field[quadIdx].x * field[quadIdx].x + field[quadIdx].y * field[quadIdx].y);
float mag = field[quadIdx].x;
int map = (int)(mag/0.004* 256);
if(map > 255) { map = 255; }
if(map < 0) { map = 0; }

for(int i = 0; i < 4; i++){
colors[4*quadIdx+i] = colorMap[map];
if (field[quadIdx].x == 0)
{
colors[4*quadIdx+i] = make_float3(0.0, 0.6, 0.2);
}
if (mag < 0)
{
colors[4*quadIdx+i] = make_float3(0.0, 0.3, 0.7);
}
}

/*
for(int i = 0; i < 4; i++){
if (newVel[quadIdx].x < 0){
colors[4*quadIdx+i].x = 1.0;
}
}*/
}