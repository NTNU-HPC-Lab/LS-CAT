#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float2 JacobiFieldInstance(float2 Top, float2 Left, float2 Bot, float2 Right, float Alpha, float2 Val)
{
float2 res;
res.x = (Top.x + Left.x + Bot.x + Right.x + Alpha * Val.x) / (4 + Alpha);
res.y = (Top.y + Left.y + Bot.y + Right.y + Alpha * Val.y) / (4 + Alpha);
return res;
}
__global__ void DiffusionJacobi(float2 *__restrict__ positions, float2 *__restrict__ oldVel, float2 *__restrict__ newVel, float dt, float dr, float viscosity, unsigned int simWidth, unsigned int simHeight)
{
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

float2 Vel = oldVel[y*simWidth + x];
float2 TVel;
float2 LVel;
float2 BVel;
float2 RVel;
float alpha = dr * dr / (viscosity * dt);

if (x!=0 && y!=0 && x!=simWidth-1 && y!=simHeight-1)
{

TVel = oldVel[(y-1)*simWidth + x];
LVel = oldVel[(y*simWidth) + x - 1];
BVel = oldVel[(y+1)*simWidth + x];
RVel = oldVel[(y*simWidth) + x + 1];

newVel[y*simWidth + x] = JacobiFieldInstance(TVel, LVel,
BVel, RVel,
alpha, Vel);
}
}