#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateVel(float2 *__restrict__ oldVel, float2 *__restrict__ newVel, unsigned int simWidth)
{
unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
oldVel[y*simWidth+x] = newVel[y*simWidth+x];
}