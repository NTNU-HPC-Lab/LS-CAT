#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zeros(double *field, int n){
int xid = blockDim.x*blockIdx.x + threadIdx.x;

if (xid < n){
field[xid] = 0;
}

}