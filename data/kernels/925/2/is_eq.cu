#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void is_eq(bool *a, bool *b, bool *ans){
int gid = getGid3d3d();
ans[0] = true;
if (a[gid] != b[gid]){
ans[0] = false;
}
}