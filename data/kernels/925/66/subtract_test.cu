#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 subtract(double2 a, double2 b){
return {a.x-b.x, a.y-b.y};
}
__global__ void subtract_test(double2 *a, double2 *b, double2 *c){
c[0] = subtract(a[0],b[0]);
}