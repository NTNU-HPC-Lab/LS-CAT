#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void complexAbsSum(double2 *in1, double2 *in2, double *out){
int gid = getGid3d3d();
double2 temp;
temp.x = in1[gid].x + in2[gid].x;
temp.y = in1[gid].y + in2[gid].y;
out[gid] = sqrt(temp.x*temp.x + temp.y*temp.y);
}