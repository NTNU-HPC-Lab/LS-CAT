#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 mult(double2 a, double2 b){
return {a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x};
}
__device__ double2 mult(double2 a, double b){
return {a.x*b, a.y*b};
}
__global__ void mult_test(double2 *a, double b, double2 *c){
c[0] = mult(a[0],b);
}