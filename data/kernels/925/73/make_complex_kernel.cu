#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 make_complex(double in, int evolution_type){
double2 result;

switch(evolution_type){
// No change
case 0:
result.x = in;
result.y = 0;
break;
// Im. Time evolution
case 1:
result.x = exp(-in);
result.y = 0;
break;
// Real Time evolution
case 2:
result.x = cos(-in);
result.y = sin(-in);
break;
}

return result;
}
__global__ void make_complex_kernel(double *in, int *evolution_type, double2 *out){

//int id = threadIdx.x + blockIdx.x*blockDim.x;
//out[id] = make_complex(in[id], evolution_type[id]);
for (int i = 0; i < 3; ++i){
out[i] = make_complex(in[i], evolution_type[i]);
}
}