#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 add(double2 a, double2 b){
return {a.x+b.x, a.y+b.y};
}
__global__ void add_test(double2 *a, double2 *b, double2 *c){
c[0] = add(a[0],b[0]);
}