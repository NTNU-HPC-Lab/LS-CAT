#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void derive(double *data, double *out, int stride, int gsize, double dx){
int gid = getGid3d3d();
if (gid < gsize){
if (gid + stride < gsize){
out[gid] = (data[gid+stride] - data[gid])/dx;
}
else{
out[gid] = data[gid]/dx;
}
}
}