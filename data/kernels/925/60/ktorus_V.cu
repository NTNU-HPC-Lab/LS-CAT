#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void ktorus_V(double *x, double *y, double *z, double* items, double *Ax, double *Ay, double *Az, double *V){

int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
int zid = blockDim.z*blockIdx.z + threadIdx.z;

double rad = sqrt((x[xid] - items[6]) * (x[xid] - items[6])
+ (y[yid] - items[7]) * (y[yid] - items[7]))
- 0.5*items[0];
double omegaR = (items[3]*items[3] + items[4]*items[4]);
double V_tot = (2*items[5]*items[5]*(z[zid] - items[8])*(z[zid] - items[8])
+ omegaR*(rad*rad + items[12]*rad*z[zid]));
V[gid] = 0.5*items[9]*(V_tot
+ Ax[gid]*Ax[gid]
+ Ay[gid]*Ay[gid]
+ Az[gid]*Az[gid]);
}