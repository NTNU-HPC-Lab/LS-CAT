#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void simple_K(double *xp, double *yp, double *zp, double mass, double *K){

unsigned int gid = getGid3d3d();
unsigned int xid = blockDim.x*blockIdx.x + threadIdx.x;
unsigned int yid = blockDim.y*blockIdx.y + threadIdx.y;
unsigned int zid = blockDim.z*blockIdx.z + threadIdx.z;
K[gid] = (HBAR*HBAR/(2*mass))*(xp[xid]*xp[xid] + yp[yid]*yp[yid]
+ zp[zid]*zp[zid]);
}