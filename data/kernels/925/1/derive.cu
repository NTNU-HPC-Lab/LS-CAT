#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void derive(double2 *data, double2 *out, int stride, int gsize, double dx){
int gid = getGid3d3d();
if (gid < gsize){
if (gid + stride < gsize){
out[gid].x = (data[gid+stride].x - data[gid].x)/dx;
out[gid].y = (data[gid+stride].y - data[gid].y)/dx;
}
else{
out[gid].x = data[gid].x/dx;
out[gid].y = data[gid].y/dx;
}
}
}