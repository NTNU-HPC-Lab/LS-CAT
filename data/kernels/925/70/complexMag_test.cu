#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double complexMagnitude(double2 in){
return sqrt(in.x*in.x + in.y*in.y);
}
__global__ void complexMag_test(double2 *in, double *out){
out[0] = complexMagnitude(in[0]);
}