#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__device__ double2 pow(double2 a, int b){
double r = sqrt(a.x*a.x + a.y*a.y);
double theta = atan(a.y / a.x);
return{pow(r,b)*cos(b*theta),pow(r,b)*sin(b*theta)};
}
__global__ void ktorus_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc){

int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
int zid = blockDim.z*blockIdx.z + threadIdx.z;

double rad = sqrt((x[xid] - items[6]) * (x[xid] - items[6])
+ (y[yid] - items[7]) * (y[yid] - items[7]))
- 0.5*items[0];

wfc[gid].x = exp(-( pow((rad)/(items[14]*items[15]*0.5),2) +
pow((z[zid])/(items[14]*items[17]*0.5),2) ) );
wfc[gid].y = 0.0;
}