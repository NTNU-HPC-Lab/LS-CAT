#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "aux_fields.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *V = NULL;
hipMalloc(&V, XSIZE*YSIZE);
double *K = NULL;
hipMalloc(&K, XSIZE*YSIZE);
double gdt = 1;
double dt = 1;
double *Ax = NULL;
hipMalloc(&Ax, XSIZE*YSIZE);
double *Ay = NULL;
hipMalloc(&Ay, XSIZE*YSIZE);
double *Az = NULL;
hipMalloc(&Az, XSIZE*YSIZE);
double *px = NULL;
hipMalloc(&px, XSIZE*YSIZE);
double *py = NULL;
hipMalloc(&py, XSIZE*YSIZE);
double *pz = NULL;
hipMalloc(&pz, XSIZE*YSIZE);
double *pAx = NULL;
hipMalloc(&pAx, XSIZE*YSIZE);
double *pAy = NULL;
hipMalloc(&pAy, XSIZE*YSIZE);
double *pAz = NULL;
hipMalloc(&pAz, XSIZE*YSIZE);
double2 *GV = NULL;
hipMalloc(&GV, XSIZE*YSIZE);
double2 *EV = NULL;
hipMalloc(&EV, XSIZE*YSIZE);
double2 *GK = NULL;
hipMalloc(&GK, XSIZE*YSIZE);
double2 *EK = NULL;
hipMalloc(&EK, XSIZE*YSIZE);
double2 *GpAx = NULL;
hipMalloc(&GpAx, XSIZE*YSIZE);
double2 *GpAy = NULL;
hipMalloc(&GpAy, XSIZE*YSIZE);
double2 *GpAz = NULL;
hipMalloc(&GpAz, XSIZE*YSIZE);
double2 *EpAx = NULL;
hipMalloc(&EpAx, XSIZE*YSIZE);
double2 *EpAy = NULL;
hipMalloc(&EpAy, XSIZE*YSIZE);
double2 *EpAz = NULL;
hipMalloc(&EpAz, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
aux_fields<<<gridBlock,threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
aux_fields<<<gridBlock,threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
aux_fields<<<gridBlock,threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}