#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void aux_fields(double *V, double *K, double gdt, double dt, double* Ax, double *Ay, double* Az, double *px, double *py, double *pz, double* pAx, double* pAy, double* pAz, double2* GV, double2* EV, double2* GK, double2* EK, double2* GpAx, double2* GpAy, double2* GpAz, double2* EpAx, double2* EpAy, double2* EpAz){
int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
int zid = blockDim.z*blockIdx.z + threadIdx.z;

GV[gid].x = exp(-V[gid]*(gdt/(2*HBAR)));
GK[gid].x = exp(-K[gid]*(gdt/HBAR));
GV[gid].y = 0.0;
GK[gid].y = 0.0;

// Ax and Ay will be calculated here but are used only for
// debugging. They may be needed later for magnetic field calc

pAx[gid] = Ax[gid] * px[xid];
pAy[gid] = Ay[gid] * py[yid];
pAz[gid] = Az[gid] * pz[zid];

GpAx[gid].x = exp(-pAx[gid]*gdt);
GpAx[gid].y = 0;
GpAy[gid].x = exp(-pAy[gid]*gdt);
GpAy[gid].y = 0;
GpAz[gid].x = exp(-pAz[gid]*gdt);
GpAz[gid].y = 0;

EV[gid].x=cos(-V[gid]*(dt/(2*HBAR)));
EV[gid].y=sin(-V[gid]*(dt/(2*HBAR)));
EK[gid].x=cos(-K[gid]*(dt/HBAR));
EK[gid].y=sin(-K[gid]*(dt/HBAR));

EpAz[gid].x=cos(-pAz[gid]*dt);
EpAz[gid].y=sin(-pAz[gid]*dt);
EpAy[gid].x=cos(-pAy[gid]*dt);
EpAy[gid].y=sin(-pAy[gid]*dt);
EpAx[gid].x=cos(-pAx[gid]*dt);
EpAx[gid].y=sin(-pAx[gid]*dt);
}