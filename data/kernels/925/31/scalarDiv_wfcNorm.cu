#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void scalarDiv_wfcNorm(double2* in, double dr, double* pSum, double2* out){
unsigned int gid = getGid3d3d();
double2 result;
double norm = sqrt((pSum[0])*dr);
result.x = (in[gid].x/norm);
result.y = (in[gid].y/norm);
out[gid] = result;
}