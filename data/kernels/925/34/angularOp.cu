#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void angularOp(double omega, double dt, double2* wfc, double* xpyypx, double2* out){
unsigned int gid = getGid3d3d();
double2 result;
double op;
op = exp( -omega*xpyypx[gid]*dt);
result.x=wfc[gid].x*op;
result.y=wfc[gid].y*op;
out[gid]=result;
}