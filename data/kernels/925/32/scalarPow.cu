#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__device__ double2 pow(double2 a, int b){
double r = sqrt(a.x*a.x + a.y*a.y);
double theta = atan(a.y / a.x);
return{pow(r,b)*cos(b*theta),pow(r,b)*sin(b*theta)};
}
__global__ void scalarPow(double2* in, double param, double2* out){
unsigned int gid = getGid3d3d();
double2 result;
result.x = pow(in[gid].x, param);
result.y = pow(in[gid].y, param);
out[gid] = result;
}