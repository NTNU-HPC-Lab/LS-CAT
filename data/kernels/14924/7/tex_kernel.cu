#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tex_kernel(hipTextureObject_t texture_obj, int num_samples, float* output) {
unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
if (idx < num_samples) {
float u = idx / static_cast<float>(num_samples);
output[idx] = tex1D<float>(texture_obj, u);
}
}