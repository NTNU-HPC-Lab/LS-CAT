#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {

int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
int p;

float vxterm, vyterm;
float2 nvterm;
// gtidx is the domain location in x for this thread
if (gtidx < dx) {
for (p = 0; p < lb; p++) {
// fi is the domain location in y for this thread
int fi = gtidy + p;
if (fi < dy) {
int fjr = fi * pdx + gtidx;
vxterm = vx[fjr];
vyterm = vy[fjr];

// Normalize the result of the inverse FFT
float scale = 1.f / (dx * dy);
nvterm.x = vxterm * scale;
nvterm.y = vyterm * scale;

float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
*fj = nvterm;
}
} // If this thread is inside the domain in Y
} // If this thread is inside the domain in X
}