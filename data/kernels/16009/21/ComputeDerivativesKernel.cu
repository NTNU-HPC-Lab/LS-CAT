#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeDerivativesKernel(int width, int height, int stride, float* Ix, float* Iy, float* Iz, hipTextureObject_t texSource, hipTextureObject_t texTarget)
{
const int ix = threadIdx.x + blockIdx.x * blockDim.x;
const int iy = threadIdx.y + blockIdx.y * blockDim.y;


if (ix >= width || iy >= height) return;

float dx = 1.0f / (float)width;
float dy = 1.0f / (float)height;

float x = ((float)ix + 0.5f) * dx;
float y = ((float)iy + 0.5f) * dy;

float t0, t1;
// x derivative
t0 = tex2D<float>(texSource, x + 2.0f * dx, y);
t0 -= tex2D<float>(texSource, x + 1.0f * dx, y) * 8.0f;
t0 += tex2D<float>(texSource, x - 1.0f * dx, y) * 8.0f;
t0 -= tex2D<float>(texSource, x - 2.0f * dx, y);
t0 /= 12.0f;

t1 = tex2D<float>(texTarget, x + 2.0f * dx, y);
t1 -= tex2D<float>(texTarget, x + 1.0f * dx, y) * 8.0f;
t1 += tex2D<float>(texTarget, x - 1.0f * dx, y) * 8.0f;
t1 -= tex2D<float>(texTarget, x - 2.0f * dx, y);
t1 /= 12.0f;

*(((float*)((char*)Ix + stride * iy)) + ix) = (t0 + t1) * 0.5f;

// t derivative
*(((float*)((char*)Iz + stride * iy)) + ix) = tex2D<float>(texSource, x, y) - tex2D<float>(texTarget, x, y);

// y derivative
t0 = tex2D<float>(texSource, x, y + 2.0f * dy);
t0 -= tex2D<float>(texSource, x, y + 1.0f * dy) * 8.0f;
t0 += tex2D<float>(texSource, x, y - 1.0f * dy) * 8.0f;
t0 -= tex2D<float>(texSource, x, y - 2.0f * dy);
t0 /= 12.0f;

t1 = tex2D<float>(texTarget, x, y + 2.0f * dy);
t1 -= tex2D<float>(texTarget, x, y + 1.0f * dy) * 8.0f;
t1 += tex2D<float>(texTarget, x, y - 1.0f * dy) * 8.0f;
t1 -= tex2D<float>(texTarget, x, y - 2.0f * dy);
t1 /= 12.0f;

*(((float*)((char*)Iy + stride * iy)) + ix) = (t0 + t1) * 0.5f;
}