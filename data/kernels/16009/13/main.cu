#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ComputeRobustnessMask.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float3 __restrict__ *rawImgRef = NULL;
hipMalloc(&rawImgRef, XSIZE*YSIZE);
const float3 __restrict__ *rawImgMoved = NULL;
hipMalloc(&rawImgMoved, XSIZE*YSIZE);
float4 __restrict__ *robustnessMask = NULL;
hipMalloc(&robustnessMask, XSIZE*YSIZE);
hipTextureObject_t texUV = 1;
int imgWidth = XSIZE;
int imgHeight = YSIZE;
int imgPitch = 2;
int maskPitch = 2;
float alpha = 2;
float beta = 2;
float thresholdM = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ComputeRobustnessMask<<<gridBlock,threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ComputeRobustnessMask<<<gridBlock,threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ComputeRobustnessMask<<<gridBlock,threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}