#include "hip/hip_runtime.h"
#include "includes.h"
//	Copyright (c) 2020, Michael Kunz. All rights reserved.
//	https://github.com/kunzmi/ImageStackAlignator
//
//	This file is part of ImageStackAlignator.
//
//	ImageStackAlignator is free software: you can redistribute it and/or modify
//	it under the terms of the GNU Lesser General Public License as
//	published by the Free Software Foundation, version 3.
//
//	ImageStackAlignator is distributed in the hope that it will be useful,
//	but WITHOUT ANY WARRANTY; without even the implied warranty of
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//	GNU Lesser General Public License for more details.
//
//	You should have received a copy of the GNU Lesser General Public
//	License along with this library; if not, write to the Free Software
//	Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
//	MA 02110-1301  USA, http://www.gnu.org/licenses/.




//squared sum of a tile without the border
extern "C"

//Boxfilter ignoring the border parts
//blockDim.X must be tileSize + 2 * maxShift
//blockDim.Y must be 1
extern "C"
//Boxfilter ignoring the border parts
//blockDim.Y must be tileSize + 2 * maxShift
//blockDim.X must be 1
extern "C"


//Computed the normalized CC values out of the different input data
//Cross correlation is fft shifted
//blockDim.X must be 2 * maxShift
//blockDim.Y must be 2 * maxShift
//blockDim.Z must be nr of tiles
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift is filled by zero
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift on each side
extern "C"

__global__ void ComputeKernelParam( float3* __restrict__ kernelImg, int imgWidth, int imgHeight, int imgOutPitch, float Dth, float Dtr, float kDetail, float kDenoise, float kStretch, float kShrink)
{
int pxX = blockIdx.x * blockDim.x + threadIdx.x;
int pxY = blockIdx.y * blockDim.y + threadIdx.y;

if (pxX >= imgWidth || pxY >= imgHeight)
return;

float3 grad = *(((float3*)((char*)kernelImg + imgOutPitch * pxY)) + pxX);
float a11 = grad.x;
float a22 = grad.y;
float a12 = grad.z;

float help = sqrtf((a22 - a11) * (a22 - a11) + 4.0f * a12 * a12);
float c = 2.0f * a12;
float s = a22 - a11 + help;

float norm = sqrtf(c * c + s * s);
if (norm > 0)
{
c /= norm;
s /= norm;
}
else
{
c = 1;
s = 0;
}

float lam1 = (a11 + a22 + help) / 2.0f;
float lam2 = (a11 + a22 - help) / 2.0f;


float A = 1 + sqrtf((lam1 - lam2) * (lam1 - lam2) / ((lam1 + lam2) * (lam1 + lam2)));
float D = 1 - sqrtf(lam1) / Dtr + Dth;

D = fmaxf(fminf(1.0f, D), 0.0f);

float k1h = kDetail * kStretch * A;
float k2h = kDetail / kShrink * A;

float k1 = ((1.0f - D)*k1h + D*kDetail*kDenoise);
float k2 = ((1.0f - D)*k2h + D*kDetail*kDenoise);
k1 *= k1;
k2 *= k2;

float x2 = c;
float y2 = s;
float x1 = s;
float y1 = -c;

float b11 = k1*x1*x1 + x2*x2*k2;
float b12 = k1*x1*y1 + x2*y2*k2;
float b22 = k1*y1*y1 + y2*y2*k2;

float det = b11*b22 - b12*b12 + 0.0000000001f;

float3 kernel;
kernel.x = b22 / det;
kernel.y = b11 / det;
kernel.z = -b12 / det;
*(((float3*)((char*)kernelImg + imgOutPitch * pxY)) + pxX) = kernel;
}