#include "hip/hip_runtime.h"
#include "includes.h"
//	Copyright (c) 2020, Michael Kunz. All rights reserved.
//	https://github.com/kunzmi/ImageStackAlignator
//
//	This file is part of ImageStackAlignator.
//
//	ImageStackAlignator is free software: you can redistribute it and/or modify
//	it under the terms of the GNU Lesser General Public License as
//	published by the Free Software Foundation, version 3.
//
//	ImageStackAlignator is distributed in the hope that it will be useful,
//	but WITHOUT ANY WARRANTY; without even the implied warranty of
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//	GNU Lesser General Public License for more details.
//
//	You should have received a copy of the GNU Lesser General Public
//	License along with this library; if not, write to the Free Software
//	Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
//	MA 02110-1301  USA, http://www.gnu.org/licenses/.




//squared sum of a tile without the border
extern "C"

//Boxfilter ignoring the border parts
//blockDim.X must be tileSize + 2 * maxShift
//blockDim.Y must be 1
extern "C"
//Boxfilter ignoring the border parts
//blockDim.Y must be tileSize + 2 * maxShift
//blockDim.X must be 1
extern "C"


//Computed the normalized CC values out of the different input data
//Cross correlation is fft shifted
//blockDim.X must be 2 * maxShift
//blockDim.Y must be 2 * maxShift
//blockDim.Z must be nr of tiles
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift is filled by zero
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift on each side
extern "C"

__global__ void conjugateComplexMulKernel(const float2* __restrict__ aIn, float2* __restrict__ bInOut, int maxElem)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= maxElem)
return;

float2 valA = aIn[idx];
//conjugate complex
valA.y = -valA.y;

//multiplication:
float2 valB = bInOut[idx];
float2 res;
res.x = valA.x * valB.x - valA.y * valB.y;
res.y = valA.x * valB.y + valA.y * valB.x;
bInOut[idx] = res;
}