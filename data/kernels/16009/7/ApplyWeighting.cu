#include "hip/hip_runtime.h"
#include "includes.h"
//	Copyright (c) 2020, Michael Kunz. All rights reserved.
//	https://github.com/kunzmi/ImageStackAlignator
//
//	This file is part of ImageStackAlignator.
//
//	ImageStackAlignator is free software: you can redistribute it and/or modify
//	it under the terms of the GNU Lesser General Public License as
//	published by the Free Software Foundation, version 3.
//
//	ImageStackAlignator is distributed in the hope that it will be useful,
//	but WITHOUT ANY WARRANTY; without even the implied warranty of
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//	GNU Lesser General Public License for more details.
//
//	You should have received a copy of the GNU Lesser General Public
//	License along with this library; if not, write to the Free Software
//	Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
//	MA 02110-1301  USA, http://www.gnu.org/licenses/.




//squared sum of a tile without the border
extern "C"

//Boxfilter ignoring the border parts
//blockDim.X must be tileSize + 2 * maxShift
//blockDim.Y must be 1
extern "C"
//Boxfilter ignoring the border parts
//blockDim.Y must be tileSize + 2 * maxShift
//blockDim.X must be 1
extern "C"


//Computed the normalized CC values out of the different input data
//Cross correlation is fft shifted
//blockDim.X must be 2 * maxShift
//blockDim.Y must be 2 * maxShift
//blockDim.Z must be nr of tiles
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift is filled by zero
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift on each side
extern "C"

__global__ void ApplyWeighting( float3 * __restrict__ inOutImg, const float3 * __restrict__ finalImg, const float3 * __restrict__ weight, int imgWidth, int imgHeight, int imgPitch, float threshold)
{
int pxX = blockIdx.x * blockDim.x + threadIdx.x;
int pxY = blockIdx.y * blockDim.y + threadIdx.y;

if (pxX >= imgWidth || pxY >= imgHeight)
return;


float3 inout = *(((float3*)((char*)inOutImg + imgPitch * pxY)) + pxX);
float3 val = *(((float3*)((char*)finalImg + imgPitch * pxY)) + pxX);
float3 w = *(((float3*)((char*)weight + imgPitch * pxY)) + pxX);


if (w.x < threshold)
{
val.x += inout.x;
w.x += 1;
}
inout.x = 0;
if (w.x != 0)
{
inout.x = val.x / w.x;
}

if (w.y < threshold)
{
val.y += inout.y;
w.y += 1;
}
inout.y = 0;
if (w.y != 0)
{
inout.y = val.y / w.y;
}

if (w.z < threshold)
{
val.z += inout.z;
w.z += 1;
}
inout.z = 0;
if (w.z != 0)
{
inout.z = val.z / w.z;
}

*(((float3*)((char*)inOutImg + imgPitch * pxY)) + pxX) = inout;
}