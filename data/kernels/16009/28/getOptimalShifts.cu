#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getOptimalShifts( float2 * __restrict__ optimalShifts, const float2 * __restrict__ bestShifts, int imageCount, int tileCountX, int tileCountY, int optimalShiftsPitch, int referenceImage, int imageToTrack)
{
int tileIdxX = blockIdx.x * blockDim.x + threadIdx.x;
int tileIdxY = blockIdx.y * blockDim.y + threadIdx.y;

if (tileIdxX >= tileCountX || tileIdxY >= tileCountY)
return;

int n1 = imageCount - 1;

const float2* r = &bestShifts[(tileIdxX + tileIdxY * tileCountX) * n1];

float2 totalShift = make_float2(0, 0);
if (referenceImage < imageToTrack)
{
for (int i = referenceImage; i < imageToTrack; i++)
{
totalShift.x += r[i].x;
totalShift.y += r[i].y;
}
}
else if(imageToTrack < referenceImage)
{
for (int i = imageToTrack; i < referenceImage; i++)
{
totalShift.x -= r[i].x;
totalShift.y -= r[i].y;
}
}

*(((float2*)((char*)(optimalShifts) +optimalShiftsPitch * tileIdxY)) + tileIdxX) = totalShift;
}