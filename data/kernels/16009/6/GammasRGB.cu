#include "hip/hip_runtime.h"
#include "includes.h"
//	Copyright (c) 2020, Michael Kunz. All rights reserved.
//	https://github.com/kunzmi/ImageStackAlignator
//
//	This file is part of ImageStackAlignator.
//
//	ImageStackAlignator is free software: you can redistribute it and/or modify
//	it under the terms of the GNU Lesser General Public License as
//	published by the Free Software Foundation, version 3.
//
//	ImageStackAlignator is distributed in the hope that it will be useful,
//	but WITHOUT ANY WARRANTY; without even the implied warranty of
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//	GNU Lesser General Public License for more details.
//
//	You should have received a copy of the GNU Lesser General Public
//	License along with this library; if not, write to the Free Software
//	Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
//	MA 02110-1301  USA, http://www.gnu.org/licenses/.




//squared sum of a tile without the border
extern "C"

//Boxfilter ignoring the border parts
//blockDim.X must be tileSize + 2 * maxShift
//blockDim.Y must be 1
extern "C"
//Boxfilter ignoring the border parts
//blockDim.Y must be tileSize + 2 * maxShift
//blockDim.X must be 1
extern "C"


//Computed the normalized CC values out of the different input data
//Cross correlation is fft shifted
//blockDim.X must be 2 * maxShift
//blockDim.Y must be 2 * maxShift
//blockDim.Z must be nr of tiles
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift is filled by zero
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift on each side
extern "C"

__device__ float applysRGBGamma(float valIn)
{
if (valIn <= 0.0031308f)
{
return 12.92f * valIn;
}
else
{
return (1.0f + 0.055f) * powf(valIn, 1.0f / 2.4f) - 0.055f;
}
}
__global__ void GammasRGB( float3 * __restrict__ inOutImg, int imgWidth, int imgHeight, int imgPitch)
{
int pxX = blockIdx.x * blockDim.x + threadIdx.x;
int pxY = blockIdx.y * blockDim.y + threadIdx.y;

if (pxX >= imgWidth || pxY >= imgHeight)
return;

float3 val = *(((float3*)((char*)inOutImg + imgPitch * pxY)) + pxX);
//apply gamma:
if (isnan(val.x))
val.x = 0;
if (isnan(val.y))
val.y = 0;
if (isnan(val.z))
val.z = 0;

val.x = fmaxf(fminf(val.x, 1.0f), 0.0f);
val.y = fmaxf(fminf(val.y, 1.0f), 0.0f);
val.z = fmaxf(fminf(val.z, 1.0f), 0.0f);

val.x = applysRGBGamma(val.x);
val.y = applysRGBGamma(val.y);
val.z = applysRGBGamma(val.z);
*(((float3*)((char*)inOutImg + imgPitch * pxY)) + pxX) = val;
}