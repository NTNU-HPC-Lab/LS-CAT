#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepareAntidiagonalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam3, float* lam5, float* lam6, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
uint32_t weightsIndex = x + w * y;

z[index] = (weights[weightsIndex] * in[index] + 2 * mu * (u[index] + v[index] + w_[index])
+ 2 * (lam3[index] + lam5[index] + lam6[index])) / weightsPrime[weightsIndex];

}
}