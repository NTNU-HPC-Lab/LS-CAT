#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepareVerticalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam1, float* lam4, float* lam5, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
uint32_t weightsIndex = x + w * y;

v[index] = (weights[weightsIndex] * in[index] + 2 * mu * (u[index] + w_[index] + z[index])
+ 2 * (lam1[index] - lam4[index] - lam5[index])) / weightsPrime[weightsIndex];

}
}