#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepareHorizontalPottsProblems4ADMM(float* in, float* u, float* v, float* weights, float* weightsPrime, float* lam, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
uint32_t weightsIndex = x + w * y;

u[index] = (weights[weightsIndex] * in[index] + v[index] * mu - lam[index]) / weightsPrime[weightsIndex];

}
}