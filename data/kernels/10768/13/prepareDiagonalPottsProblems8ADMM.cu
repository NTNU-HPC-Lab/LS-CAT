#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepareDiagonalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam2, float* lam4, float* lam6, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
uint32_t weightsIndex = x + w * y;

w_[index] = (weights[weightsIndex] * in[index] + 2 * mu * (u[index] + v[index] + z[index])
+ 2 * (lam2[index] + lam4[index] - lam6[index])) / weightsPrime[weightsIndex];

}
}