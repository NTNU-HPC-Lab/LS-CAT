#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepareHorizontalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam1, float* lam2, float* lam3, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
uint32_t weightsIndex = x + w * y;

u[index] = (weights[weightsIndex] * in[index] + 2 * mu * (v[index] + w_[index] + z[index])
+ 2 * (-lam1[index] - lam2[index] - lam3[index])) / weightsPrime[weightsIndex];

}
}