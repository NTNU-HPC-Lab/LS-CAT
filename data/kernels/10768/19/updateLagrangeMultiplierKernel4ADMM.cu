#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateLagrangeMultiplierKernel4ADMM(float* u, float* v, float* lam, float* temp, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
temp[index] = u[index] - v[index];
lam[index] = lam[index] + temp[index] * mu;
}
}