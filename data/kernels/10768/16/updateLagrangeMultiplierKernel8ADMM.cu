#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateLagrangeMultiplierKernel8ADMM(float* u, float* v, float* w_, float* z, float* lam1, float* lam2, float* lam3, float* lam4, float* lam5, float* lam6, float* temp, float mu, uint32_t w, uint32_t h, uint32_t nc) {
uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

if(x < w && y < h && c < nc) {
uint32_t index = x + w * y + w * h * c;
temp[index] = u[index] - v[index];
lam1[index] = lam1[index] + mu * (u[index] - v[index]);
lam2[index] = lam2[index] + mu * (u[index] - w_[index]);
lam3[index] = lam3[index] + mu * (u[index] - z[index]);
lam4[index] = lam4[index] + mu * (v[index] - w_[index]);
lam5[index] = lam5[index] + mu * (v[index] - z[index]);
lam6[index] = lam6[index] + mu * (w_[index] - z[index]);
}
}