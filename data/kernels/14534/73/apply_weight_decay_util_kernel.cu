#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_weight_decay_util_kernel( const float4 * __restrict learning_rates, float4 * __restrict weights, float weight_decay, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val = learning_rates[elem_id];
float4 current_weight = weights[elem_id];
val.x = 1.0F - val.x * weight_decay;
val.y = 1.0F - val.y * weight_decay;
val.z = 1.0F - val.z * weight_decay;
val.w = 1.0F - val.w * weight_decay;
current_weight.x *= val.x;
current_weight.y *= val.y;
current_weight.z *= val.z;
current_weight.w *= val.w;
weights[elem_id] = current_weight;
}
}