#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel( float4 * __restrict output_buf, const float4 * __restrict input_buf1, const float4 * __restrict input_buf2, float alpha1, float alpha2, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val1 = input_buf1[elem_id];
float4 val2 = input_buf2[elem_id];
val1.x = val1.x * alpha1 + val2.x * alpha2;
val1.y = val1.y * alpha1 + val2.y * alpha2;
val1.z = val1.z * alpha1 + val2.z * alpha2;
val1.w = val1.w * alpha1 + val2.w * alpha2;
output_buf[elem_id] = val1;
}
}