#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectified_linear_kernel( float4 * __restrict output, const float4 * __restrict input, float negative_slope, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val = input[elem_id];
if (val.x < 0.0F)
val.x *= negative_slope;
if (val.y < 0.0F)
val.y *= negative_slope;
if (val.z < 0.0F)
val.z *= negative_slope;
if (val.w < 0.0F)
val.w *= negative_slope;
output[elem_id] = val;
}
}