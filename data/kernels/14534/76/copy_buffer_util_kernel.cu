#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_buffer_util_kernel( const float4 * __restrict input_buf, float4 * __restrict output_buf, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
output_buf[elem_id] = input_buf[elem_id];
}