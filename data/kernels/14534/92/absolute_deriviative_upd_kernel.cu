#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void absolute_deriviative_upd_kernel( float4 * __restrict input_errors, const float4 * __restrict output_errors, const float4 * __restrict input_neurons, bool add_update_to_destination, int elem_count)
{
int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
if (elem_id < elem_count)
{
float4 inp = input_neurons[elem_id];
float4 current_error = output_errors[elem_id];
if (inp.x < 0.0F)
current_error.x = -current_error.x;
if (inp.y < 0.0F)
current_error.y = -current_error.y;
if (inp.z < 0.0F)
current_error.z = -current_error.z;
if (inp.w < 0.0F)
current_error.w = -current_error.w;
float4 current_dst;
if (add_update_to_destination)
{
current_dst = input_errors[elem_id];
current_error.x += current_dst.x;
current_error.y += current_dst.y;
current_error.z += current_dst.z;
current_error.w += current_dst.w;
}
input_errors[elem_id] = current_error;
}
}