#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_with_value_util_kernel( int4 * __restrict buf, int v, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
int4 val;
val.x = v;
val.y = v;
val.z = v;
val.w = v;
buf[elem_id] = val;
}
}