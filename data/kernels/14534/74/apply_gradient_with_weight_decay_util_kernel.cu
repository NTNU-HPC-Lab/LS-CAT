#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_gradient_with_weight_decay_util_kernel( const float2 * __restrict gradient, const float2 * __restrict learning_rates, float2 * __restrict weights, float weight_decay, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float2 lr = learning_rates[elem_id];
float2 current_weight = weights[elem_id];
float2 grad = gradient[elem_id];
float2 new_weight;
new_weight.x = current_weight.x + lr.x * (grad.x - weight_decay * current_weight.x);
new_weight.y = current_weight.y + lr.y * (grad.y - weight_decay * current_weight.y);
weights[elem_id] = new_weight;
}
}