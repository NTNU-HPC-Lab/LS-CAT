#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_with_value_util_kernel( float4 * __restrict buf, float v, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val;
val.x = v;
val.y = v;
val.z = v;
val.w = v;
buf[elem_id] = val;
}
}