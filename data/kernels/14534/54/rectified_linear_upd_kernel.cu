#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectified_linear_upd_kernel( const float4 * __restrict input, float4 * __restrict output, uint4 * __restrict bits_buffer, float negative_slope, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
float4 val;
uint4 bits;
if (elem_id < elem_count)
val = input[elem_id];

#ifdef __CUDACC_VER_MAJOR__
#if __CUDACC_VER_MAJOR__ < 9
bits.x = __ballot(val.x < 0.0F ? 0 : 1);
bits.y = __ballot(val.y < 0.0F ? 0 : 1);
bits.z = __ballot(val.z < 0.0F ? 0 : 1);
bits.w = __ballot(val.w < 0.0F ? 0 : 1);
#else
bits.x = __ballot_sync(0xFFFFFFFF, val.x < 0.0F ? 0 : 1);
bits.y = __ballot_sync(0xFFFFFFFF, val.y < 0.0F ? 0 : 1);
bits.z = __ballot_sync(0xFFFFFFFF, val.z < 0.0F ? 0 : 1);
bits.w = __ballot_sync(0xFFFFFFFF, val.w < 0.0F ? 0 : 1);
#endif
#endif

if (elem_id < elem_count)
{
int lane_id = elem_id & 31;
if (lane_id == 0)
bits_buffer[elem_id >> 5] = bits;
if (val.x < 0.0F)
val.x *= negative_slope;
if (val.y < 0.0F)
val.y *= negative_slope;
if (val.z < 0.0F)
val.z *= negative_slope;
if (val.w < 0.0F)
val.w *= negative_slope;
output[elem_id] = val;
}
}