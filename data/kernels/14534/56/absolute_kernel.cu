#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void absolute_kernel( float4 * __restrict output, const float4 * __restrict input, int elem_count)
{
int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
if (elem_id < elem_count)
{
float4 val = input[elem_id];
val.x = fabsf(val.x);
val.y = fabsf(val.y);
val.z = fabsf(val.z);
val.w = fabsf(val.w);
output[elem_id] = val;
}
}