#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectified_linear_backprop_upd_kernel( float4 * __restrict input_errors, const float4 * __restrict output_errors, const uint4 * __restrict bits_buffer, float negative_slope, bool add_update_to_destination, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val = output_errors[elem_id];
uint4 bits = bits_buffer[elem_id >> 5];
int lane_id = elem_id & 31;
unsigned int mask = (1 << lane_id);
if ((bits.x & mask) == 0)
val.x *= negative_slope;
if ((bits.y & mask) == 0)
val.y *= negative_slope;
if ((bits.z & mask) == 0)
val.z *= negative_slope;
if ((bits.w & mask) == 0)
val.w *= negative_slope;
if (add_update_to_destination)
{
float4 prv = input_errors[elem_id];
val.x += prv.x;
val.y += prv.y;
val.z += prv.z;
val.w += prv.w;
}
input_errors[elem_id] = val;
}
}