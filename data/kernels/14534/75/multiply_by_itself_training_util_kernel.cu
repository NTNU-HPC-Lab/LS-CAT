#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_by_itself_training_util_kernel( const float4 * __restrict input_buf, float4 * __restrict output_buf, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
float4 val = input_buf[elem_id];
val.x *= val.x;
val.y *= val.y;
val.z *= val.z;
val.w *= val.w;
output_buf[elem_id] = val;
}
}