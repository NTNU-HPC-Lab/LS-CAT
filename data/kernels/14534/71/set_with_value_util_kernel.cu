#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_with_value_util_kernel( double2 * __restrict buf, double v, int elem_count)
{
int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
if (elem_id < elem_count)
{
double2 val;
val.x = v;
val.y = v;
buf[elem_id] = val;
}
}