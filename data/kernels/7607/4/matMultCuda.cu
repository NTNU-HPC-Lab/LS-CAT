#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMultCuda(float *cu_C, float *cu_A, float *cu_B, unsigned int n) {

int row = (blockIdx.x * blockDim.x) + threadIdx.x;
int col = (blockIdx.y * blockDim.y) + threadIdx.y;

//Log row and col of each thread
//printf("row : %d , col : %d \n", row, col);

if (row < n && col < n) {
int temp_sum = 0;

for (int elem = 0; elem < n; elem++)
{
temp_sum += cu_A[row * n + elem] * cu_B[elem * n + col];
}

cu_C[row * n + col] = temp_sum;
}
};