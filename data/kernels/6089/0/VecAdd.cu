#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 2018 International Business Machines
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/



#define LOG_ERR(pid, fmt, x...) fprintf(stderr, "Process %d: " fmt, pid, ##x)
#define LOG_INF(pid, fmt, x...) printf("Process %d: " fmt, pid, ##x)

#define AFU_NAME "IBM,MEMCPY3"
#define AFU_MAX_PROCESSES 512

#define CACHELINESIZE	128
/* Queue sizes other than 512kB don't seem to work (still true?) */
#define QUEUE_SIZE	4095*CACHELINESIZE

#define MEMCPY_WED(queue, depth)			\
((((uint64_t)queue) & 0xfffffffffffff000ULL) |	\
(((uint64_t)depth) & 0xfffULL))

#define MEMCPY_WE_CMD(valid, cmd)		\
(((valid) & 0x1) |			\
(((cmd) & 0x3f) << 2))
#define MEMCPY_WE_CMD_VALID	(0x1 << 0)
#define MEMCPY_WE_CMD_WRAP	(0x1 << 1)
#define MEMCPY_WE_CMD_COPY		0
#define MEMCPY_WE_CMD_IRQ		1
#define MEMCPY_WE_CMD_STOP		2
#define MEMCPY_WE_CMD_WAKE_HOST_THREAD	3
#define MEMCPY_WE_CMD_INCREMENT		4
#define MEMCPY_WE_CMD_ATOMIC		5
#define MEMCPY_WE_CMD_TRANSLATE_TOUCH	6

/* global mmio registers */
#define MEMCPY_AFU_GLOBAL_CFG	0
#define MEMCPY_AFU_GLOBAL_TRACE	0x20

/* per-process mmio registers */
#define MEMCPY_AFU_PP_WED	0
#define MEMCPY_AFU_PP_STATUS	0x10
#define   MEMCPY_AFU_PP_STATUS_Terminated	0x8
#define   MEMCPY_AFU_PP_STATUS_Stopped		0x10

#define MEMCPY_AFU_PP_CTRL	0x18
#define   MEMCPY_AFU_PP_CTRL_Restart	(0x1 << 0)
#define   MEMCPY_AFU_PP_CTRL_Terminate	(0x1 << 1)
#define MEMCPY_AFU_PP_IRQ	0x28


struct memcpy_work_element {
volatile uint8_t cmd; /* valid, wrap, cmd */
volatile uint8_t status;
uint16_t length;
uint8_t cmd_extra;
uint8_t reserved[3];
uint64_t atomic_op;
uint64_t src;  /* also irq EA or atomic_op2 */
uint64_t dst;
} __packed;

struct memcpy_weq {
struct memcpy_work_element *queue;
struct memcpy_work_element *next;
struct memcpy_work_element *last;
int wrap;
int count;
};

struct memcpy_test_args {
int loop_count;
int size;
int irq;
int completion_timeout;
int reallocate;
char *device;
int wake_host_thread;
int increment;
int atomic_cas;
/* global vars */
int shmid;
char *lock;
char *counter;
};

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
for (int i=0; i<N; i++) {
C[i] = A[i] + B[i];
}
}