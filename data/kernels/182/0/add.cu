#include "hip/hip_runtime.h"
#include "includes.h"


#define N 128*256
#define THREADS_PER_BLOCK 256
#define N_BLOCKS N/THREADS_PER_BLOCK

// Kernel to add N integers using threads and blocks

// Main program
__global__ void add(int *a, int *b, int *c){
int index = blockIdx.x * blockDim.x + threadIdx.x;

c[index] = a[index] + b[index];
}