#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot( int *a, int *b, int *c ) {

__shared__ int prod[THREADS_PER_BLOCK]; // Shared memory
int index = blockIdx.x * blockDim.x + threadIdx.x;

prod[threadIdx.x] = a[index] * b[index];

__syncthreads();  // Threads synchronization

if( threadIdx.x == 0) {
int par_sum = 0;

for(int i=0; i<THREADS_PER_BLOCK; i++)
par_sum += prod[threadIdx.x]; // Threads reduction

atomicAdd(c,par_sum); // Blocks reduction
}
}