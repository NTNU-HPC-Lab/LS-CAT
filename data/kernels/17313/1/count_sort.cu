#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_sort(int *x, int *y, int size){

int idx = blockIdx.x * blockDim.x + threadIdx.x;

if(idx < size){

int count = 0;

for(int j = 0; j < size; j++){

if (x[j] < x[idx])
count++;
else if (x[j] == x[idx] && j < idx)
count++;

}

y[count] = x[idx];
}

}