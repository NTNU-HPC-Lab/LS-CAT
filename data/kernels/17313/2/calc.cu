#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc(float *result, float *b, float *a, int size){

int idx = blockIdx.x * blockDim.x + threadIdx.x;

if(idx < size){

float temp;

for (int j = 0; j < size; j++){
temp = *(a + j + (idx * size)) * (*(b + j));
atomicAdd(&result[idx], temp);
}
}
}