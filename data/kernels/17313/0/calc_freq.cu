#include "hip/hip_runtime.h"
#include "includes.h"

#define N 128


__global__ void calc_freq(int *freq, int file_size, char *buffer, int total_threads){
int temp[N];
int idx = blockIdx.x * blockDim.x + threadIdx.x;

// Initialize temp with value 0
for (int i = 0; i < N; i++){
temp[i] = 0;
}

// Do the calculation
for(int i = idx; i < file_size; i += total_threads) {
temp[buffer[i]]++;
}

// Add the results from the threads to the blocks
for(int i = 0; i < N; i++){
atomicAdd(&freq[i], temp[i]);
}

}