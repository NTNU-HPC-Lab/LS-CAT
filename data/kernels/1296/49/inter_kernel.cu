#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *out)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < (NX+NY)*B){
int b = i / (NX+NY);
int j = i % (NX+NY);
if (j < NX){
out[i] = X[b*NX + j];
} else {
out[i] = Y[b*NY + j - NX];
}
}
}