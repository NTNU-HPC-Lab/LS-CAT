#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
*(output + blockIdx.x * N + i) = -row_sum[blockIdx.x];
}
}