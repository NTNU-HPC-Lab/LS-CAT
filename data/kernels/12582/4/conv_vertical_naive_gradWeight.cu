#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_vertical_naive_gradWeight(const int n, float *y, const float *x, const int kL, const int iC)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
y[i] = x[(i/kL)*kL*iC + i];
}
}