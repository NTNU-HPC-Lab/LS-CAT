#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_vertical_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int oH = iH - kL + 1;
int x_offset = (i/(oH*iW))*iH*iW + i%(oH*iW);
int w_offset = (i/(oH*iW))*kL;

for (int k = 0; k < kL; k++) {
y[i] += w[w_offset + k]*x[x_offset + k*iW];
}
}
}