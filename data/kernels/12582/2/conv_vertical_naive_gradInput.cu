#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_vertical_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int iH = oH + kL - 1;
int iC = i/(iH*oW);
int row = (i%(iH*oW))/oW;
int dy_offset = iC*oH*oW + i%(iH*oW);
int w_offset = iC*kL;

int k_begin = max(0, row-oH+1);
int k_end = min(kL, row+1);

dx[i] = 0.0f;
for (int k = k_begin; k < k_end; k++) {
dx[i] += w[w_offset + k]*dy[dy_offset - k*oW];
}
}
}