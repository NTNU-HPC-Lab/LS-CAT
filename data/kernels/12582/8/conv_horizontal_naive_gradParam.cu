#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
int iW = oW + kL - 1;
int dy_offset = (i/kL)*oH*oW;
int x_offset = (i/kL)*oH*oW + i%kL;

for (int j = 0; j < oH; j++) {
for (int k = 0; k < oW; k++) {
dw[i] += dy[dy_offset + j*oW + k]*x[x_offset + j*iW + k];
}
}
}
}