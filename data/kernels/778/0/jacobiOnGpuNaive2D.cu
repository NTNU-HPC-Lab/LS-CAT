#include "hip/hip_runtime.h"
#include "includes.h"
#define DEBUG 0

__global__ void jacobiOnGpuNaive2D(double *MatA, double *d_MatC, int dim_x, int dim_y,int iter_max){
unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;
unsigned long long int j = blockDim.y * blockIdx.y + threadIdx.y * 4;
unsigned long long int iPrev = i-1;
unsigned long long int iNext = i+1;
unsigned long long int jPrev = j-1;
unsigned long long int jNext = j+1;
unsigned long long int index = i * dim_y + j;
unsigned long long int indexUnroll3 = index - 1;
unsigned long long int indexUnroll2 = index - 2;
unsigned long long int indexUnroll1 = index - 3;
for (int k = 0; k < iter_max; k++){
if( i > 0 && j > 0 && i < (dim_x-1) && j <(dim_y-1)){
d_MatC[index] = 0.25f * (MatA[iPrev * dim_y + j] + MatA[iNext* dim_y + j] + MatA[i * dim_y+ jPrev] + MatA[i* dim_y + jNext]);
d_MatC[indexUnroll1] = 0.25f * (MatA[indexUnroll1 + 1] + MatA[indexUnroll1 - 1] + MatA[indexUnroll1 + dim_x] + MatA[indexUnroll1 - dim_x]);
d_MatC[indexUnroll2] = 0.25f * (MatA[indexUnroll2 + 1] + MatA[indexUnroll2 - 1] + MatA[indexUnroll2 + dim_x] + MatA[indexUnroll2 - dim_x]);
d_MatC[indexUnroll3] = 0.25f * (MatA[indexUnroll3 + 1] + MatA[indexUnroll3 - 1] + MatA[indexUnroll3 + dim_x] + MatA[indexUnroll3 - dim_x]);
if(DEBUG){
printf("index %llu %llu %llu %llu\n", index, indexUnroll3, indexUnroll2, indexUnroll1);
//printf("index %llu \n", index);
//printf("indexUnroll2 %llu, contornos %lf %lf %lf %lf \n",indexUnroll2, MatA[indexUnroll2 + 1], MatA[indexUnroll2 - 1], MatA[indexUnroll2 + dim_y] , MatA[indexUnroll2 - dim_y] );
}
}
__syncthreads();
if(index<dim_x*dim_y) MatA[index] = d_MatC[index];
}
}