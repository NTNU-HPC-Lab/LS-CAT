#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( int *a, int *b, int *c ) {

int idx = threadIdx.x + blockIdx.x * blockDim.x;

if (idx < N) {

int idx1 = (idx + 1) % 256;
int idx2 = (idx + 2) % 256;
float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
c[idx] = (as + bs) / 2;

}

}