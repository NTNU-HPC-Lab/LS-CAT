#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeGPU1d( unsigned char *image1, unsigned char *image2, unsigned char *res, int pixels ) {

int i = threadIdx.x + blockIdx.x*blockDim.x;

if( i < pixels ) {

int idx = 3*i;
int r1 = image1[ idx+2 ];
int g1 = image1[ idx+1 ];
int b1 = image1[ idx   ];
int r2 = image2[ idx+2 ];
int g2 = image2[ idx+1 ];
int b2 = image2[ idx   ];
int r = (int)( ( (float)r1 + (float)r2 )*0.5f );
int g = (int)( ( (float)g1 + (float)g2 )*0.5f );
int b = (int)( ( (float)b1 + (float)b2 )*0.5f );
res[ idx+2 ] = (unsigned char)r;
res[ idx+1 ] = (unsigned char)g;
res[ idx   ] = (unsigned char)b;

}
}