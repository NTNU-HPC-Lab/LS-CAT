#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cinzaGPU1d( unsigned char *image1, unsigned char *res, int pixels ) {

int i = threadIdx.x + blockIdx.x*blockDim.x;
int cinza;

if( i < pixels ) {

int idx = 3*i;
int r = image1[ idx+2 ];
int g = image1[ idx+1 ];
int b = image1[ idx   ];

cinza  = (30*r + 59*g + 11*b)/100;

res[ idx+2 ] = (unsigned char)cinza;
res[ idx+1 ] = (unsigned char)cinza;
res[ idx   ] = (unsigned char)cinza;

}
}