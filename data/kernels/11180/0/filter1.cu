#include "hip/hip_runtime.h"
#include "includes.h"
//-----------------------------------------
// Autor: Farias
// Data : January 2012
// Goal : Image treatment
//-----------------------------------------

/***************************************************************************************************
Includes
***************************************************************************************************/




/***************************************************************************************************
Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) (i+(j)*(DIMX_))
#define BLOCK_SIZE 16


/***************************************************************************************************
Functions
***************************************************************************************************/

using namespace std;


/**************************************************************************************************/

__global__ void filter1( int width, int height, unsigned char *src, unsigned char *dest ) {

int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

int aux, idx;

if(i > 0 && j > 0 && i < width - 1 && j < height - 1) {
for (int k = 0; k < 3; ++k)
{
aux = 0;
idx = 3*ELEM( i, j, width );

aux += 4*src[ idx+k ];

idx = 3*ELEM( i-1, j, width );
aux+= 2*src[ idx+k ];

idx = 3*ELEM( i, j-1, width );
aux+= 2*src[ idx+k ];

idx = 3*ELEM( i+1, j, width );
aux+= 2*src[ idx+k ];

idx = 3*ELEM( i, j+1, width );
aux+= 2*src[ idx+k ];

aux /= 12;

idx = 3*ELEM( i, j, width );
dest[ idx+k ] = (unsigned char)aux;

}

}
}