#include "hip/hip_runtime.h"
#include "includes.h"
//-----------------------------------------
// Autor: Farias
// Data : January 2012
// Goal : Image treatment
//-----------------------------------------

/***************************************************************************************************
Includes
***************************************************************************************************/




/***************************************************************************************************
Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) (i+(j)*(DIMX_))
#define BLOCK_SIZE 16


/***************************************************************************************************
Functions
***************************************************************************************************/

using namespace std;


/**************************************************************************************************/

__global__ void filter2( int width, int height, unsigned char *src, unsigned char *dest ) {

int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

int aux, idx;

__shared__ int pesos[3][3];

// Setando Pesos
pesos[0][0] = 0; pesos[0][1] = 2; pesos[0][2] = 0;
pesos[1][0] = 2; pesos[1][1] = 4; pesos[1][2] = 2;
pesos[2][0] = 0; pesos[2][1] = 2; pesos[2][2] = 0;



if(i > 0 && j > 0 && i < width - 1 && j < height - 1) {
for (int k = 0; k < 3; ++k)
{

aux = 0;
for (int lin = 0; lin < 3; lin++)
{
for (int col = 0; col < 3; col++){
idx = 3*ELEM( i + lin - 1, j + col - 1, width );
aux += pesos[lin][col]*src[ idx+k ];
}
}
aux /= 12;
idx = 3*ELEM( i, j , width );
dest[ idx+k ] = (unsigned char)aux;

}

}
}