#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void kernelGPU(float *R,float* G,float* B,float* Rin,float*Gin,float*Bin,int M,int N,int L){

int tId= threadIdx.x+blockIdx.x*blockDim.x;
int i;
if(tId<M*N){
R[tId]=0;
G[tId]=0;
B[tId]=0;
for(i=0; i<L; ++i ){

R[tId]+= Rin[tId+i*M*N];
G[tId]+= Gin[tId+i*M*N];
B[tId]+= Bin[tId+i*M*N];
}

R[tId]=R[tId]/L;
G[tId]=G[tId]/L;
B[tId]=B[tId]/L;
}


}