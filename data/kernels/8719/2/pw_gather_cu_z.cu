#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_gather_cu_z(      double *pwcc, const double *c, const double  scale, const int     ngpts, const int    *ghatmap) {

const int igpt = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

if (igpt < ngpts) {
pwcc[2 * igpt    ] = scale * c[2 * ghatmap[igpt]    ];
pwcc[2 * igpt + 1] = scale * c[2 * ghatmap[igpt] + 1];
}
}