#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_vec_mult_fixed_dims(int *mat, int *vec, int *res) {
int mat_rows = 1024;
int mat_cols = 512;
// El for each thread, shared per block
__shared__ int smem[128];
for (int block_i = 0; block_i * gridDim.x < mat_rows; block_i++) {
int row = blockIdx.x + (block_i * gridDim.x);
int row_total = 0;
for (int thread_i = 0; thread_i * blockDim.x < mat_cols; thread_i++) {
int col = threadIdx.x + (thread_i * blockDim.x);
// Load mult in shmem
smem[threadIdx.x] = mat[row * mat_cols + col] * vec[col];
__syncthreads();

// Parallel reduction
for (int i = blockDim.x / 2; i > 0; i /= 2) {
if (threadIdx.x < i) {
int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
smem[threadIdx.x] = temp;
}
__syncthreads();
}
// Only 1 thread needs to do this
if (threadIdx.x == 0)
row_total += smem[threadIdx.x];
}
// Load into ans (single thread)
if (threadIdx.x == 0)
res[row] = row_total;
}
}