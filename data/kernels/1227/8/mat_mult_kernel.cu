#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mult_kernel(int *mat_a, int *mat_b, int *result, int a_rows, int a_cols, int b_cols) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < a_rows) {
for (int j = 0; j < b_cols; j++) {
int temp_res = 0;
for (int k = 0; k < a_cols; k++) {
temp_res  += mat_a[tid * a_cols + k] * mat_b[k * b_cols + j];
}
result[tid * b_cols + j] = temp_res;
}
tid += blockDim.x * gridDim.x;
}
}