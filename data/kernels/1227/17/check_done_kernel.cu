#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_done_kernel(bool *mask, int num_vtx, bool *finished) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
while (*finished && tid < num_vtx) {
if (mask[tid])
*finished = false;
tid += blockDim.x * gridDim.x;
}

}