#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void update_dists_kernel(const int * beg_pos, const int * adj_list, const int* weights, bool * mask, int* dists, int* update_dists, const int num_vtx) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_vtx) {
if (dists[tid] > update_dists[tid]) {
dists[tid] = update_dists[tid];
mask[tid] = true;
}
update_dists[tid] = dists[tid];
}
}