#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mult_kernel(int *a, int *b, int *c, int mat_rows, int mat_cols) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < mat_rows) {
int res = 0;
for (int i = 0; i < mat_cols; i++) {
res += a[tid * mat_cols + i] * b[i];
}
c[tid] = res;
tid += blockDim.x * gridDim.x;
}
}