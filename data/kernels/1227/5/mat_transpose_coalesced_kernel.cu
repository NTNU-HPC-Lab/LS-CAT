#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_transpose_coalesced_kernel(int *mat, int *res) {
// Square tile
int tile_dim = 32;
// 32 Blocks across for 1024 mat
int blocks_per_row = 32;

__shared__ int smem[32 * 32];

int rows_per_block_iter = 64;
// Each iter has 2 "block-rows"
for (int block_iter = 0; block_iter < 16; block_iter++) {
int tile_row = blockIdx.x / blocks_per_row;
int tile_col = blockIdx.x % blocks_per_row;

int intile_row = threadIdx.x / tile_dim;
int intile_col = threadIdx.x % tile_dim;

int read_row = (tile_row * tile_dim) + intile_row + (rows_per_block_iter * block_iter);
int read_col = (tile_col * tile_dim) + intile_col;

int write_row = (tile_col * tile_dim) + intile_row;
int write_col = (tile_row * tile_dim) + intile_col + (rows_per_block_iter * block_iter);


smem[(intile_row * tile_dim) + intile_col] = mat[(read_row * 1024) + read_col];
__syncthreads();
res[(write_row * 1024) + write_col] = smem[(intile_col * tile_dim) + intile_row];
}
}