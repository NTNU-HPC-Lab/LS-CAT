#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void block_sum_kernel(int *arr, int size, int *block_sums) {
int num_threads = blockDim.x * gridDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;

// Each thread finds local sum of its assigned area
int my_sum = 0;
__shared__ int smem[128];
while (tid < size) {
my_sum += arr[tid];
tid += num_threads;
}

smem[threadIdx.x] = my_sum;

// Barrier then use parallel reduction to get block sum
__syncthreads();
for (int i = blockDim.x / 2; i > 0; i /= 2) {
if (threadIdx.x < i) {
int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
smem[threadIdx.x] = temp;
}
__syncthreads();
}
// Block sum added to global arr
if (threadIdx.x == 0) {
block_sums[blockIdx.x] = smem[0];
}
}