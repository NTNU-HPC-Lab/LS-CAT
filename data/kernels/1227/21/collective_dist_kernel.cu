#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double get_collective_dist(int *dist, int rows, int cols, int col) {
double sum = 0;
for (int i = 0; i < rows; i++) {
if (dist[i * cols + col] == 0) {
return 0;
}
sum += (1 / (double)dist[i * cols + col]);
}
return sum;
}
__global__ void collective_dist_kernel(int *dist, int rows, int cols, double *col_dist)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < cols) {
col_dist[tid] = get_collective_dist(dist, rows, cols, tid);
tid += blockDim.x * gridDim.x;
}
}