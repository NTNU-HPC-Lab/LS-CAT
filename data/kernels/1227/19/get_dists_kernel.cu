#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void get_dists_kernel(const int * beg_pos, const int* adj_list, const int * weights, bool * mask, int* dists, int * update_dists, const int num_vtx) {

int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid < num_vtx) {
if (mask[tid] == true) {
mask[tid] = false;
for (int edge = beg_pos[tid]; edge < beg_pos[tid + 1]; edge++) {
int other = adj_list[edge];
atomicMin(&update_dists[other],
dists[tid] + weights[edge]);
}
}
}
}