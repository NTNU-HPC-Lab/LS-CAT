#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_transpose_regular_kernel(int *mat, int *res) {
// Square tile
int tile_dim = 32;
// 32 Blocks across for 1024 mat
int blocks_per_row = 32;

int rows_per_block_iter = 64;
// Each iter has 2 "block-rows"
for (int block_iter = 0; block_iter < 16; block_iter++) {
int tile_row = blockIdx.x / blocks_per_row;
int tile_col = blockIdx.x % blocks_per_row;

int intile_row = threadIdx.x / tile_dim;
int intile_col = threadIdx.x % tile_dim;

int my_row = (tile_row * tile_dim) + intile_row + (rows_per_block_iter * block_iter);
int my_col = (tile_col * tile_dim) + intile_col;

res[(my_col * 1024) + my_row] = mat[(my_row * 1024) + my_col];
}
}