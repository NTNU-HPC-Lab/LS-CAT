#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_naive_kernel(int *arr, int size, int *sum) {
int num_threads = blockDim.x * gridDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < size) {
atomicAdd(sum, arr[tid]);
tid += num_threads;
}
}