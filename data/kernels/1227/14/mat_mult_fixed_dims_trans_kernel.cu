#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mult_fixed_dims_trans_kernel(int *mat_a, int *mat_b, int *res) {
int B_TRANS_ROWS = B_COLS;
int B_TRANS_COLS = B_ROWS;
// El for each thread, shared per block
__shared__ int smem[128];
for (int row_block = 0; row_block * gridDim.x < A_ROWS; row_block++) {

int a_row = blockIdx.x + (row_block * gridDim.x);
for (int b_row = 0; b_row < B_TRANS_ROWS; b_row++) {

int total = 0;
for (int thread_i = 0; thread_i * blockDim.x < A_COLS; thread_i++) {

int thread_col = threadIdx.x + (thread_i * blockDim.x);
// Need to check because 240 not even multiple of 128
if (thread_col >= A_COLS)
smem[threadIdx.x] = 0;
else
smem[threadIdx.x] = mat_a[a_row * A_COLS + thread_col] * mat_b[b_row * B_TRANS_COLS + thread_col];
__syncthreads();

//Parallel reduction
for (int i = blockDim.x / 2; i > 0; i /= 2) {
if (threadIdx.x < i) {
int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
smem[threadIdx.x] = temp;
}
__syncthreads();
}
if (threadIdx.x == 0) {
total += smem[threadIdx.x];
}
}
if (threadIdx.x == 0) {
res[a_row * C_COLS + b_row] = total;
}
}
}
}