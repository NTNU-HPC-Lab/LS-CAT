#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_improved_atomic_kernel(int *arr, int size, int *sum) {
int num_threads = blockDim.x * gridDim.x;
int tid = threadIdx.x + blockIdx.x * blockDim.x;

// Each thread finds local sum of its assigned area
int my_sum = 0;
__shared__ int smem[128];

while (tid < size) {
my_sum += arr[tid];
tid += num_threads;
}

// Load local sum into shared mem
smem[threadIdx.x] = my_sum;

// Barrier then use parallel reduction to get block sum
__syncthreads();
for (int i = blockDim.x / 2; i > 0; i /= 2) {
if (threadIdx.x < i)
smem[threadIdx.x] += smem[threadIdx.x + i];
__syncthreads();
}
// Block sum atomically added to global sum
if (threadIdx.x == 0) {
atomicAdd(sum, smem[0]);
}
}