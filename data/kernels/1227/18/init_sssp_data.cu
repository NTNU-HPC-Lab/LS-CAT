#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_sssp_data(bool * d_mask, int* d_dists, int* d_update_dists, const int source, const int num_vtx) {

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < num_vtx) {
if (source == tid) {
d_mask[tid] = true;
d_dists[tid] = 0;
d_update_dists[tid] = 0;
}
else {
d_mask[tid] = false;
d_dists[tid] = INT_MAX;
d_update_dists[tid] = INT_MAX;
}
}
}