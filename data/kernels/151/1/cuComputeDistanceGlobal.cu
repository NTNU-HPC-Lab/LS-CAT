#include "hip/hip_runtime.h"
#include "includes.h"
/**
*
* Date         03/07/2009
* ====
*
* Authors      Vincent Garcia
* =======      Eric    Debreuve
*              Michel  Barlaud
*
* Description  Given a reference point set and a query point set, the program returns
* ===========  the distance between each query point and its k-th nearest neighbor in
*              the reference point set. Only the distance is provided. The computation
*              is performed using the API NVIDIA CUDA.
*
* Paper        Fast k nearest neighbor search using GPU
* =====
*
* BibTeX       @INPROCEEDINGS{2008_garcia_cvgpu,
* ======         author = {V. Garcia and E. Debreuve and M. Barlaud},
*                title = {Fast k nearest neighbor search using GPU},
*                booktitle = {CVPR Workshop on Computer Vision on GPU},
*                year = {2008},
*                address = {Anchorage, Alaska, USA},
*                month = {June}
*              }
*
*/


// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0


// Includes
#if MATLAB_CODE == 1
#else
#endif


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;



//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//



/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
* The matrix A is a texture.
*
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param pA    pitch of matrix A given in number of columns
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/



/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist     distance matrix
* @param width    width of the distance matrix
* @param pitch    pitch of the distance matrix given in number of columns
* @param height   height of the distance matrix
* @param k        number of smallest distance to consider
*/



/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
*/



//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//



/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void cuComputeDistanceGlobal( float* A, int wA, int pA, float* B, int wB, int pB, int dim,  float* AB){

// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

// Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
__shared__ int begin_A;
__shared__ int begin_B;
__shared__ int step_A;
__shared__ int step_B;
__shared__ int end_A;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Other variables
float tmp;
float ssd = 0;

// Loop parameters
begin_A = BLOCK_DIM * blockIdx.y;
begin_B = BLOCK_DIM * blockIdx.x;
step_A  = BLOCK_DIM * pA;
step_B  = BLOCK_DIM * pB;
end_A   = begin_A + (dim-1) * pA;

// Conditions
int cond0 = (begin_A + tx < wA); // used to write in shared memory
int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
if (a/pA + ty < dim){
shared_A[ty][tx] = (cond0)? A[a + pA * ty + tx] : 0;
shared_B[ty][tx] = (cond1)? B[b + pB * ty + tx] : 0;
}
else{
shared_A[ty][tx] = 0;
shared_B[ty][tx] = 0;
}

// Synchronize to make sure the matrices are loaded
__syncthreads();

// Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
if (cond2 && cond1){
for (int k = 0; k < BLOCK_DIM; ++k){
tmp = shared_A[k][ty] - shared_B[k][tx];
ssd += tmp*tmp;
}
}

// Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write the block sub-matrix to device memory; each thread writes one element
if (cond2 && cond1)
AB[ (begin_A + ty) * pB + begin_B + tx ] = ssd;
}