#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
if (xIndex<width){
float sum = 0.f;
for (int i=0; i<height; i++){
float val = array[i*pitch+xIndex];
sum += val*val;
}
norm[xIndex] = sum;
}
}