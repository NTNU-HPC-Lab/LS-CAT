#include "hip/hip_runtime.h"
#include "includes.h"
/**
*
* Date         03/07/2009
* ====
*
* Authors      Vincent Garcia
* =======      Eric    Debreuve
*              Michel  Barlaud
*
* Description  Given a reference point set and a query point set, the program returns
* ===========  the distance between each query point and its k-th nearest neighbor in
*              the reference point set. Only the distance is provided. The computation
*              is performed using the API NVIDIA CUDA.
*
* Paper        Fast k nearest neighbor search using GPU
* =====
*
* BibTeX       @INPROCEEDINGS{2008_garcia_cvgpu,
* ======         author = {V. Garcia and E. Debreuve and M. Barlaud},
*                title = {Fast k nearest neighbor search using GPU},
*                booktitle = {CVPR Workshop on Computer Vision on GPU},
*                year = {2008},
*                address = {Anchorage, Alaska, USA},
*                month = {June}
*              }
*
*/


// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0


// Includes
#if MATLAB_CODE == 1
#else
#endif


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;



//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//



/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
* The matrix A is a texture.
*
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param pA    pitch of matrix A given in number of columns
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/



/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist     distance matrix
* @param width    width of the distance matrix
* @param pitch    pitch of the distance matrix given in number of columns
* @param height   height of the distance matrix
* @param k        number of smallest distance to consider
*/



/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
*/



//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//



/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void cuInsertionSort(float *dist, int width, int pitch, int height, int k){

// Variables
int l,i,j;
float *p;
float v, max_value;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){

// Pointer shift and max value
p         = dist+xIndex;
max_value = *p;

// Part 1 : sort kth firt element
for (l=pitch;l<k*pitch;l+=pitch){
v = *(p+l);
if (v<max_value){
i=0; while (i<l && *(p+i)<=v) i+=pitch;
for (j=l;j>i;j-=pitch)
*(p+j) = *(p+j-pitch);
*(p+i) = v;
}
max_value = *(p+l);
}

// Part 2 : insert element in the k-th first lines
for (l=k*pitch;l<height*pitch;l+=pitch){
v = *(p+l);
if (v<max_value){
i=0; while (i<k*pitch && *(p+i)<=v) i+=pitch;
for (j=(k-1)*pitch;j>i;j-=pitch)
*(p+j) = *(p+j-pitch);
*(p+i) = v;
max_value  = *(p+(k-1)*pitch);
}
}
}
}