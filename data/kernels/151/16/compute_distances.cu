#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_distances(float * ref, int     ref_width, int     ref_pitch, float * query, int     query_width, int     query_pitch, int     height, float * dist) {

// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

// Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
__shared__ int begin_A;
__shared__ int begin_B;
__shared__ int step_A;
__shared__ int step_B;
__shared__ int end_A;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Initializarion of the SSD for the current thread
float ssd = 0.f;

// Loop parameters
begin_A = BLOCK_DIM * blockIdx.y;
begin_B = BLOCK_DIM * blockIdx.x;
step_A  = BLOCK_DIM * ref_pitch;
step_B  = BLOCK_DIM * query_pitch;
end_A   = begin_A + (height-1) * ref_pitch;

// Conditions
int cond0 = (begin_A + tx < ref_width); // used to write in shared memory
int cond1 = (begin_B + tx < query_width); // used to write in shared memory & to computations and to write in output array
int cond2 = (begin_A + ty < ref_width); // used to computations and to write in output matrix

// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
if (a/ref_pitch + ty < height) {
shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
}
else {
shared_A[ty][tx] = 0;
shared_B[ty][tx] = 0;
}

// Synchronize to make sure the matrices are loaded
__syncthreads();

// Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
if (cond2 && cond1) {
for (int k = 0; k < BLOCK_DIM; ++k){
float tmp = shared_A[k][ty] - shared_B[k][tx];
ssd += tmp*tmp;
}
}

// Synchronize to make sure that the preceeding computation is done before loading two new sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write the block sub-matrix to device memory; each thread writes one element
if (cond2 && cond1) {
dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
}
}