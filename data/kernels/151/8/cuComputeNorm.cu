#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
if (xIndex<width){
float val, sum=0;
int i;
for (i=0;i<height;i++){
val  = mat[i*pitch+xIndex];
sum += val*val;
}
norm[xIndex] = sum;
}
}