#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuInsertionSort(float *dist, int width, int pitch, int height, int k){

// Variables
int l,i,j;
float *p;
float v, max_value;
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (xIndex<width){

// Pointer shift and max value
p         = dist+xIndex;
max_value = *p;

// Part 1 : sort kth firt element
for (l=pitch;l<k*pitch;l+=pitch){
v = *(p+l);
if (v<max_value){
i=0; while (i<l && *(p+i)<=v) i+=pitch;
for (j=l;j>i;j-=pitch)
*(p+j) = *(p+j-pitch);
*(p+i) = v;
}
max_value = *(p+l);
}

// Part 2 : insert element in the k-th first lines
for (l=k*pitch;l<height*pitch;l+=pitch){
v = *(p+l);
if (v<max_value){
i=0; while (i<k*pitch && *(p+i)<=v) i+=pitch;
for (j=(k-1)*pitch;j>i;j-=pitch)
*(p+j) = *(p+j-pitch);
*(p+i) = v;
max_value  = *(p+(k-1)*pitch);
}
}
}
}