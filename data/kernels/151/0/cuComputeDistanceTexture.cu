#include "hip/hip_runtime.h"
#include "includes.h"
/**
*
* Date         03/07/2009
* ====
*
* Authors      Vincent Garcia
* =======      Eric    Debreuve
*              Michel  Barlaud
*
* Description  Given a reference point set and a query point set, the program returns
* ===========  the distance between each query point and its k-th nearest neighbor in
*              the reference point set. Only the distance is provided. The computation
*              is performed using the API NVIDIA CUDA.
*
* Paper        Fast k nearest neighbor search using GPU
* =====
*
* BibTeX       @INPROCEEDINGS{2008_garcia_cvgpu,
* ======         author = {V. Garcia and E. Debreuve and M. Barlaud},
*                title = {Fast k nearest neighbor search using GPU},
*                booktitle = {CVPR Workshop on Computer Vision on GPU},
*                year = {2008},
*                address = {Anchorage, Alaska, USA},
*                month = {June}
*              }
*
*/


// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0


// Includes
#if MATLAB_CODE == 1
#else
#endif


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;



//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//



/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
* The matrix A is a texture.
*
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param pA    pitch of matrix A given in number of columns
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/



/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist     distance matrix
* @param width    width of the distance matrix
* @param pitch    pitch of the distance matrix given in number of columns
* @param height   height of the distance matrix
* @param k        number of smallest distance to consider
*/



/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
*/



//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//



/**
* Prints the error message return during the memory allocation.
*
* @param error        error value return by the memory allocation function
* @param memorySize   size of memory tried to be allocated
*/
__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if ( xIndex<wB && yIndex<wA ){
float ssd = 0;
for (int i=0; i<dim; i++){
float tmp  = tex2D(texA, (float)yIndex, (float)i) - B[ i * pB + xIndex ];
ssd += tmp * tmp;
}
AB[yIndex * pB + xIndex] = ssd;
}
}