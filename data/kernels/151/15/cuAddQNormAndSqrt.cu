#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAddQNormAndSqrt(float *dist, int width, int pitch, float *q, int k){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex] + q[xIndex]);
}