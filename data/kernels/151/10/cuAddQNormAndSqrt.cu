#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAddQNormAndSqrt(float *vec1,  float *vec2, int width){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
if (xIndex<width){
vec1[xIndex] = sqrt(vec1[xIndex]+vec2[xIndex]);
}
}