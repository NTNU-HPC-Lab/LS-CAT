#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void StarRadKernel (double *Qbase2, double *Vrad, double *QStar, double dt, int nrad, int nsec, double *invdiffRmed, double *Rmed, double *dq)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dqm, dqp;

if (i<nrad && j<nsec){
if ((i == 0 || i == nrad-1)) dq[i + j*nrad] = 0.0;
else {
dqm = (Qbase2[i*nsec + j] - Qbase2[(i-1)*nsec + j])*invdiffRmed[i];
dqp = (Qbase2[(i+1)*nsec + j] - Qbase2[i*nsec + j])*invdiffRmed[i+1];

if (dqp * dqm > 0.0)
dq[i+j*nrad] = 2.0*dqp*dqm/(dqp+dqm);
else
dq[i+j*nrad] = 0.0;
}
}
}