#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyPolarGridbyConstantKernel (double *Dens, int nrad, int nsec, double ScalingFactor)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<=nrad && j<nsec)
Dens[i*nsec + j] *= ScalingFactor;
}