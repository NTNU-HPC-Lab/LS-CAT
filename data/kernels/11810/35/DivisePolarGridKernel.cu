#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DivisePolarGridKernel (double *Qbase, double *DensInt, double *Work, int nrad, int nsec)
{
int i = threadIdx.x + blockDim.x*blockIdx.x; //512
int j = threadIdx.y + blockDim.y*blockIdx.y; //256

if (i<=nsec && j<nrad)
Work[i*nrad + j] = Qbase[i*nrad + j]/(DensInt[i*nrad + j] + 1e-20);
}