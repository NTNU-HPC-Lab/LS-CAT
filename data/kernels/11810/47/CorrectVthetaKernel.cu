#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CorrectVthetaKernel (double *Vtheta, double domega, double *Rmed, int nrad, int nsec)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec)
Vtheta[i*nsec + j] = Vtheta[i*nsec + j] - domega*Rmed[i];
}