#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceCsKernel (double *SoundSpeed, double *cs0, double *cs1, double *csnrm1, double *csnrm2, int nsec, int nrad)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i=0;

if(j<nsec){
cs0[j] = SoundSpeed[i*nsec +j];
cs1[j] = SoundSpeed[(i+1)*nsec +j];
}
i = nrad-1;
if(j<nsec){
csnrm2[j] = SoundSpeed[(i-1)*nsec +j];
csnrm1[j] = SoundSpeed[i*nsec +j];
}
}