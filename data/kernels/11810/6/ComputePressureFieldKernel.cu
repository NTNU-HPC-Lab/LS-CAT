#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputePressureFieldKernel (double *SoundSpeed, double *Dens, double *Pressure, int Adiabatic, int nrad, int nsec, double ADIABATICINDEX, double *Energy) /* LISTO */
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
if (!Adiabatic)
Pressure[i*nsec + j] = Dens[i*nsec + j]*SoundSpeed[i*nsec + j]*SoundSpeed[i*nsec + j];

/* Since SoundSpeed is not update from initialization, cs remains axisymmetric*/
else Pressure[i*nsec + j] = (ADIABATICINDEX-1.0)*Energy[i*nsec + j];
}
}