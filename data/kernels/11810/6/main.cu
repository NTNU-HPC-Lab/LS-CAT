#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ComputePressureFieldKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *SoundSpeed = NULL;
hipMalloc(&SoundSpeed, XSIZE*YSIZE);
double *Dens = NULL;
hipMalloc(&Dens, XSIZE*YSIZE);
double *Pressure = NULL;
hipMalloc(&Pressure, XSIZE*YSIZE);
int Adiabatic = 1;
int nrad = 1;
int nsec = 1;
double ADIABATICINDEX = 1;
double *Energy = NULL;
hipMalloc(&Energy, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ComputePressureFieldKernel<<<gridBlock,threadBlock>>>(SoundSpeed,Dens,Pressure,Adiabatic,nrad,nsec,ADIABATICINDEX,Energy);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ComputePressureFieldKernel<<<gridBlock,threadBlock>>>(SoundSpeed,Dens,Pressure,Adiabatic,nrad,nsec,ADIABATICINDEX,Energy);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ComputePressureFieldKernel<<<gridBlock,threadBlock>>>(SoundSpeed,Dens,Pressure,Adiabatic,nrad,nsec,ADIABATICINDEX,Energy);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}