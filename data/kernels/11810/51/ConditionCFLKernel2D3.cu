#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConditionCFLKernel2D3 (double *newDT, double *DT2D, double *DT1D, double *Vmoy, double *invRmed, int *CFL, int nsec, int nrad, double DeltaT)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;

double newdt;
if (j == 0){
newdt = newDT[1];
for (int i=2; i<nrad; i++){
if (newDT[i] < newdt)
newdt = newDT[i];
}

for (int i = 0; i < nrad-1; i++) {
if (DT1D[i] < newdt)
newdt = DT1D[i];
}

if (DeltaT < newdt)
newdt = DeltaT;
CFL[0] = (int)(ceil(DeltaT/newdt));
}
}