#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeConstantResidualKernel (double *VMed, double *invRmed, int *Nshift, int *NoSplitAdvection, int nsec, int nrad, double dt, double *Vtheta, double *VthetaRes, double *Rmed, int FastTransport)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double maxfrac, Ntilde, Nround, invdt, dpinvns;
long nitemp;

if (i<nrad && j<nsec){
if (FastTransport)
maxfrac = 1.0;
else
maxfrac = 0.0;

invdt = 1.0/dt;
dpinvns = 2.0*PI/(double)nsec;
Ntilde = VMed[i]*invRmed[i]*dt*(double)nsec/2.0/PI;
Nround = floor(Ntilde+0.5);
nitemp = (long)Nround;
Nshift[i] = (long)nitemp;

Vtheta[i*nsec + j] = (Ntilde-Nround)*Rmed[i]*invdt*dpinvns;
if (maxfrac < 0.5){
NoSplitAdvection[i] = YES;
VthetaRes[i*nsec + j] += Vtheta[i*nsec + j];
Vtheta[i*nsec + j] = 0.0;
}
else{
NoSplitAdvection[i] = NO;
}
}
}