#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MinusMeanKernel (double *Dens, double *Energy, double SigmaMed, double mean_dens_r, double mean_dens_r2, double mean_energy_r,double mean_energy_r2, double EnergyMed, int nsec, int nrad, double SigmaMed2, double EnergyMed2)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = 0;
if (j< nsec){
Dens[i*nsec + j] += SigmaMed - mean_dens_r;
Energy[i*nsec + j] += EnergyMed - mean_energy_r;
}

i = nrad-1;
if (j < nsec){
Dens[i*nsec + j] += SigmaMed2 - mean_dens_r2;
Energy[i*nsec + j] += EnergyMed2 - mean_energy_r2;
}
}