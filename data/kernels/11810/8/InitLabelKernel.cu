#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitLabelKernel (double *Label, double xp, double yp, double rhill, double *Rmed, int nrad, int nsec)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
double distance, angle, x, y;
angle = (double)j / (double)nsec*2.0*PI;
x = Rmed[i] * cos(angle);
y = Rmed[i] * sin(angle);
distance = sqrt((x - xp) * (x - xp) + (y - yp)*(y -yp));

if (distance < rhill) Label[i*nsec + j] = 1.0;
else Label[i*nsec + j] = 0.0;

}
}