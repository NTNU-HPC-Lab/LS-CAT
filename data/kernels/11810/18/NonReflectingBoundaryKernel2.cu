#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NonReflectingBoundaryKernel2 (double *Dens, double *Energy, int i_angle, int nsec, double *Vrad, double *SoundSpeed, double SigmaMed, int nrad, double SigmaMed2, int i_angle2)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = 1;
double Vrad_med;

Vrad_med = -SoundSpeed[i*nsec + j]*(Dens[i*nsec + j]-SigmaMed)/SigmaMed;
Vrad[i*nsec + j] = 2.0*Vrad_med-Vrad[(i+1)*nsec + j];
i = nrad-1;

Vrad_med = SoundSpeed[i*nsec + j]*(Dens[(i-1)*nsec + j]-SigmaMed2)/SigmaMed2;
Vrad[i*nsec + j] = 2.*Vrad_med - Vrad[(i-1)*nsec + j];

}