#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplySubKeplerianBoundaryKernel(double *VthetaInt, double *Rmed, double OmegaFrame, int nsec, int nrad, double VKepIn, double VKepOut)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = 0;

if (j<nsec)
VthetaInt[i*nsec + j] = VKepIn - Rmed[i]*OmegaFrame;

i = nrad - 1;

if (j<nsec)
VthetaInt[i*nsec + j] =  VKepOut - Rmed[i]*OmegaFrame;

}