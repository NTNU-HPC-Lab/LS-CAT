#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceMeanKernel (double *Dens, double *Energy, int nsec, double *mean_dens, double *mean_energy, double *mean_dens2, double *mean_energy2, int nrad)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = 0;

if(j<nsec){
mean_dens[j] = Dens[i*nsec+ j];
mean_energy[j] = Energy[i*nsec +j];
}
i = nrad-1;
if(j<nsec){
mean_dens2[j] = Dens[i*nsec + j];
mean_energy2[j] = Energy[i*nsec + j];
}
}