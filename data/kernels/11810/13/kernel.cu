#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(double *Dens, double *VradInt, double *VthetaInt, double *TemperInt, int nrad, int nsec, double *invdiffRmed, double *invdiffRsup, double *DensInt, int Adiabatic, double *Rmed, double dt, double *VradNew, double *VthetaNew, double *Energy, double *EnergyInt)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dens, densint, dxtheta, invdxtheta, dens2, tempint;
if (i>0 && i<nrad && j<nsec){

dens = Dens[i*nsec + j] + Dens[(i-1)*nsec + j];
densint = DensInt[i*nsec+j] - DensInt[(i-1)*nsec + j];

VradNew[i*nsec+j] = VradInt[i*nsec+j] - dt*2.0/dens*densint*invdiffRmed[i];
}

if (i<nrad && j<nsec){
dxtheta = 2.0*PI/(double)nsec*Rmed[i];
invdxtheta = 1.0/dxtheta;

dens2 = Dens[i*nsec + j] + Dens[i*nsec + ((j-1)+nsec)%nsec];
tempint = (TemperInt[i*nsec+j] - TemperInt[i*nsec + ((j-1)+nsec)%nsec]);

VthetaNew[i*nsec + j] = VthetaInt[i*nsec + j] - dt*2.0/dens2*tempint*invdxtheta;

}


/* If gas disk is adiabatic, we add artificial viscosity as a source */
/* term for advection of thermal energy polargrid */
if (Adiabatic){
if (i<nrad && j<nsec){
dxtheta = 2.0*PI/(double)nsec*Rmed[i];
invdxtheta = 1.0/dxtheta;

EnergyInt[i*nsec + j] = Energy[i*nsec + j] - dt*DensInt[i*nsec + j]* \
(VradInt[(i+1)*nsec + j] - VradInt[i*nsec + j])*invdiffRsup[i] - \
dt*TemperInt[i*nsec + j]*(VthetaInt[i*nsec + (j+1)%nsec] - VthetaInt[i*nsec + j])* invdxtheta;
}
}

}