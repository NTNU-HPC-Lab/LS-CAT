#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *Dens = NULL;
hipMalloc(&Dens, XSIZE*YSIZE);
double *VradInt = NULL;
hipMalloc(&VradInt, XSIZE*YSIZE);
double *VthetaInt = NULL;
hipMalloc(&VthetaInt, XSIZE*YSIZE);
double *TemperInt = NULL;
hipMalloc(&TemperInt, XSIZE*YSIZE);
int nrad = 1;
int nsec = 1;
double *invdiffRmed = NULL;
hipMalloc(&invdiffRmed, XSIZE*YSIZE);
double *invdiffRsup = NULL;
hipMalloc(&invdiffRsup, XSIZE*YSIZE);
double *DensInt = NULL;
hipMalloc(&DensInt, XSIZE*YSIZE);
int Adiabatic = 1;
double *Rmed = NULL;
hipMalloc(&Rmed, XSIZE*YSIZE);
double dt = 1;
double *VradNew = NULL;
hipMalloc(&VradNew, XSIZE*YSIZE);
double *VthetaNew = NULL;
hipMalloc(&VthetaNew, XSIZE*YSIZE);
double *Energy = NULL;
hipMalloc(&Energy, XSIZE*YSIZE);
double *EnergyInt = NULL;
hipMalloc(&EnergyInt, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel<<<gridBlock,threadBlock>>>(Dens,VradInt,VthetaInt,TemperInt,nrad,nsec,invdiffRmed,invdiffRsup,DensInt,Adiabatic,Rmed,dt,VradNew,VthetaNew,Energy,EnergyInt);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
kernel<<<gridBlock,threadBlock>>>(Dens,VradInt,VthetaInt,TemperInt,nrad,nsec,invdiffRmed,invdiffRsup,DensInt,Adiabatic,Rmed,dt,VradNew,VthetaNew,Energy,EnergyInt);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
kernel<<<gridBlock,threadBlock>>>(Dens,VradInt,VthetaInt,TemperInt,nrad,nsec,invdiffRmed,invdiffRsup,DensInt,Adiabatic,Rmed,dt,VradNew,VthetaNew,Energy,EnergyInt);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}