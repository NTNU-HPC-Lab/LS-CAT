#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeResidualsKernel (double *VthetaRes, double *VMed, int nsec, int nrad, double *Vtheta)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec)
VthetaRes[i*nsec + j] = Vtheta[i*nsec + j]-VMed[i];
}