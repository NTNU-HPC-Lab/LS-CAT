#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExtQtyKernel (double *ExtLabel, double *Dens, double *Label, int nsec, int nrad)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec)
ExtLabel[i*nsec + j] = Dens[i*nsec + j]*Label[i*nsec + j];
}