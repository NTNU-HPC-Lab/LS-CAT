#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VanLeerRadialKernel (double *Rinf, double *Rsup, double *QRStar, double *DensStar, double *Vrad, double *LostByDisk, int nsec, int nrad, double dt, int OpenInner, double *Qbase, double *invSurf)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double varq, dtheta;

if (i<nrad && j<nsec){
dtheta = 2.0*PI/(double)nsec;
varq = dt*dtheta*Rinf[i]*QRStar[i*nsec + j]* DensStar[i*nsec + j]*Vrad[i*nsec + j];
varq -= dt*dtheta*Rsup[i]*QRStar[(i+1)*nsec + j]* DensStar[(i+1)*nsec + j]*Vrad[(i+1)*nsec + j];
Qbase[i*nsec + j] += varq*invSurf[i];

if (i==0 && OpenInner)
LostByDisk[j] = varq;

}
}