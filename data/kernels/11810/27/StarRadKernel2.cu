#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void StarRadKernel2 (double *Qbase2, double *Vrad, double *QStar, double dt, int nrad, int nsec, double *invdiffRmed, double *Rmed, double *dq)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
if (Vrad[i*nsec + j] > 0.0 && i > 0)
QStar[i*nsec + j] = Qbase2[(i-1)*nsec + j] + (Rmed[i]-Rmed[i-1]-Vrad[i*nsec + j]*dt)*0.5*dq[i-1+j*nrad];
else
QStar[i*nsec + j] = Qbase2[i*nsec + j]-(Rmed[i+1]-Rmed[i]+Vrad[i*nsec + j]*dt)*0.5*dq[i+j*nrad];

}

if (i == 0 && j<nsec)
QStar[j] = QStar[j+nsec*nrad] = 0.0;
}