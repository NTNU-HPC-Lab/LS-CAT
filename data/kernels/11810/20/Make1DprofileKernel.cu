#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Make1DprofileKernel (double *gridfield, double *axifield, int nsec, int nrad)
{
int i = threadIdx.x + blockDim.x*blockIdx.x;
int j;

if (i < nrad){
double sum = 0.0;

for (j = 0; j < nsec; j++)
sum += gridfield[i*nsec + j];

axifield[i] = sum/(double)nsec;
}
}