#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Substep3Kernel2 (double *Dens, double *Qplus, double *viscosity_array, double *TAURR, double *TAURP,double *TAUPP, double *DivergenceVelocity, int nrad, int nsec, double *Rmed, int Cooling, double *EnergyNew, double dt, double *EnergyMed, double *SigmaMed, double *CoolingTimeMed, double *EnergyInt, double ADIABATICINDEX, double *QplusMed)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;
double den, num;

if (i==0 && j<nsec){
/* We calculate the heating source term Qplus for i=0 */

if (viscosity_array[nrad-1] != 0.0) {
/* power-law extrapolation */
Qplus[i*nsec + j] = Qplus[(i+1)*nsec + j]*exp(log(Qplus[(i+1)*nsec + j]/Qplus[(i+2)*nsec + j]) * \
log(Rmed[i]/Rmed[i+1]) / log(Rmed[i+1]/Rmed[i+2]));
}
else
Qplus[i*nsec + j] = 0.0;
}

/* Now we can update energy with source terms from i=0 */
if (i<nrad && j<nsec){
if (!Cooling){
num = dt*Qplus[i*nsec + j] + EnergyInt[i*nsec + j];
den = 1.0+(ADIABATICINDEX-1.0)*dt*DivergenceVelocity[i*nsec + j];
EnergyNew[i*nsec + j] = num/den;
}
else{
num = EnergyMed[i]*dt*Dens[i*nsec + j]/SigmaMed[i] + CoolingTimeMed[i]*EnergyInt[i*nsec + j] + \
dt*CoolingTimeMed[i]*(Qplus[i*nsec + j]-QplusMed[i]*Dens[i*nsec + j]/SigmaMed[i]);

den = dt + CoolingTimeMed[i] + (ADIABATICINDEX-1.0)*dt*CoolingTimeMed[i]*DivergenceVelocity[i*nsec + j];
EnergyNew[i*nsec + j] = num/den;
}
}
}