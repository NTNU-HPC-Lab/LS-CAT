#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "ConditionCFLKernel2D2.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double *newDT = NULL;
hipMalloc(&newDT, XSIZE*YSIZE);
double *DT2D = NULL;
hipMalloc(&DT2D, XSIZE*YSIZE);
double *DT1D = NULL;
hipMalloc(&DT1D, XSIZE*YSIZE);
double *Vmoy = NULL;
hipMalloc(&Vmoy, XSIZE*YSIZE);
double *invRmed = NULL;
hipMalloc(&invRmed, XSIZE*YSIZE);
int *CFL = NULL;
hipMalloc(&CFL, XSIZE*YSIZE);
int nsec = 1;
int nrad = 1;
double DeltaT = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ConditionCFLKernel2D2<<<gridBlock,threadBlock>>>(newDT,DT2D,DT1D,Vmoy,invRmed,CFL,nsec,nrad,DeltaT);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
ConditionCFLKernel2D2<<<gridBlock,threadBlock>>>(newDT,DT2D,DT1D,Vmoy,invRmed,CFL,nsec,nrad,DeltaT);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
ConditionCFLKernel2D2<<<gridBlock,threadBlock>>>(newDT,DT2D,DT1D,Vmoy,invRmed,CFL,nsec,nrad,DeltaT);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}