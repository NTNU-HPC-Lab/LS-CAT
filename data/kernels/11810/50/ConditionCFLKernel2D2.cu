#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConditionCFLKernel2D2 (double *newDT, double *DT2D, double *DT1D, double *Vmoy, double *invRmed, int *CFL, int nsec, int nrad, double DeltaT)
{
int i = threadIdx.x + blockDim.x*blockIdx.x;
int k;
double dt;
double newdt = 1e30;

if (i>0 && i<nrad){
newDT[i] = newdt;
for (k = 0; k < nsec; k++)
if (DT2D[i*nsec + k] < newDT[i]) newDT[i] = DT2D[i*nsec + k]; // for each dt in nrad
}

if (i<nrad-1){
dt = 2.0*PI*CFLSECURITY/(double)nsec/fabs(Vmoy[i]*invRmed[i]-Vmoy[i+1]*invRmed[i+1]);
DT1D[i] = dt; // array nrad size dt
}
}