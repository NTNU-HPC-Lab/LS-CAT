#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VanLeerThetaKernel (double *Rsup, double *Rinf, double *Surf, double dt, int nrad, int nsec, int UniformTransport, int *NoSplitAdvection, double *QRStar, double *DensStar, double *Vazimutal_d, double *Qbase)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dxrad, invsurf, varq;

if (i<nrad && j<nsec){
if ((UniformTransport == NO) || (NoSplitAdvection[i] == NO)){
dxrad = (Rsup[i]-Rinf[i])*dt;
invsurf = 1.0/Surf[i];
varq = dxrad*QRStar[i*nsec + j]*DensStar[i*nsec + j]*Vazimutal_d[i*nsec + j];
varq -= dxrad*QRStar[i*nsec + (j+1)%nsec]*DensStar[i*nsec + (j+1)%nsec]*Vazimutal_d[i*nsec + (j+1)%nsec];
Qbase[i*nsec + j] += varq*invsurf;
}
}
}