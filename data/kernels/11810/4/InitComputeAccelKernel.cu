#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitComputeAccelKernel (double *CellAbscissa, double *CellOrdinate, double *Rmed, int nsec, int nrad)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;


if (i<nrad && j<nsec){
CellAbscissa[i*nsec+j] = Rmed[i] * cos((2.0*PI*(double)j)/(double)nsec);
CellOrdinate[i*nsec+j] = Rmed[i] * sin((2.0*PI*(double)j)/(double)nsec);
}
}