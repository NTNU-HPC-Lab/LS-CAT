#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CrashKernel (double *array, int nrad, int nsec, int Crash)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
if (array[i*nsec + j] < 0.0)
array[i*nsec + j] = 1.0;
else
array[i*nsec + j] = 0.0;
}
}