#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Substep1Kernel (double *Pressure, double *Dens, double *VradInt, double *invdiffRmed, double *Potential, double *Rinf, double *invRinf, double *Vrad, double *VthetaInt, double *Vtheta, double *Rmed, double dt, int nrad, int nsec, double OmegaFrame, int ZMPlus, double IMPOSEDDISKDRIFT, double SIGMASLOPE)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;
double gradp, gradphi, vradint, vradint2, supp_torque, dxtheta, invdxtheta;
double vt2;

// i=1->nrad , j=0->nsec
if (i > 0 && i<nrad && j<nsec){
gradp = (Pressure[i*nsec + j] - Pressure[(i-1)*nsec + j])*2.0/(Dens[i*nsec + j] + Dens[(i-1)*nsec + j])*invdiffRmed[i];
gradphi = (Potential[i*nsec + j] - Potential[(i-1)*nsec + j])*invdiffRmed[i];
vt2 = Vtheta[i*nsec + j] + Vtheta[i*nsec + (j+1)%nsec] + Vtheta[(i-1)*nsec + j] + Vtheta[(i-1)*nsec + (j+1)%nsec];
vt2 = vt2/4.0  +OmegaFrame*Rinf[i];
vt2 = vt2*vt2;

vradint = -gradp - gradphi;
vradint2 = vradint + vt2*invRinf[i];
VradInt[i*nsec + j] = Vrad[i*nsec+j] + dt*vradint2;


}

// i=0->nrad ,   j=0->nsec
if (i<nrad && j<nsec){

supp_torque = IMPOSEDDISKDRIFT*0.5*pow(Rmed[i], -2.5+SIGMASLOPE);
dxtheta = 2.0*PI/(double)nsec*Rmed[i];
invdxtheta = 1.0/dxtheta;

gradp = (Pressure[i*nsec + j] - Pressure[i*nsec + ((j-1)+nsec)%nsec])*2.0/(Dens[i*nsec +j] +Dens[i*nsec + ((j-1)+nsec)%nsec]) \
*invdxtheta;

//if (ZMPlus) gradp *= 1; //gradp *= SG_aniso_coeff;  Definir mas adelante SG_aniso_coeff

gradphi = (Potential[i*nsec+ j] - Potential[i*nsec + ((j-1)+nsec)%nsec])*invdxtheta;
VthetaInt[i*nsec + j] =  Vtheta[i*nsec+j] - dt*(gradp+gradphi);
VthetaInt[i*nsec + j] += dt*supp_torque;
}
}