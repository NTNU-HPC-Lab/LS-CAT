#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConditionCFLKernel1D (double *Rsup, double *Rinf, double *Rmed, int nrad, int nsec, double *Vtheta, double *Vmoy)
{
int i = threadIdx.x + blockDim.x*blockIdx.x;
int j;

if (i<nrad){
Vmoy[i] = 0.0;

for (j = 0; j < nsec; j++)
Vmoy[i] += Vtheta[i*nsec + j];

Vmoy[i] /= (double)nsec;
}
}