#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OpenBoundaryKernel (double *Vrad, double *Dens, double *Energy, int nsec, double SigmaMed)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = 1;

if(j < nsec){
Dens[(i-1)*nsec + j] = Dens[i*nsec + j]; // copy first ring into ghost ring
Energy[(i-1)*nsec + j] = Energy[i*nsec + j];
if (Vrad[(i+1)*nsec + j] > 0.0 || (Dens[i*nsec + j] < SigmaMed))
Vrad[i*nsec + j] = 0.0; // we just allow outflow [inwards]
else
Vrad[i*nsec +j] = Vrad[(i+1)*nsec + j];
}
}