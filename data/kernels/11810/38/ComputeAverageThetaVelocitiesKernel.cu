#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeAverageThetaVelocitiesKernel(double *Vtheta, double *VMed, int nsec, int nrad)
{
int i = threadIdx.x + blockDim.x*blockIdx.x;

double moy = 0.0;
if (i<nrad){
for (int j = 0; j < nsec; j++)
moy += Vtheta[i*nsec + j];

VMed[i] = moy/(double)nsec;
}
}