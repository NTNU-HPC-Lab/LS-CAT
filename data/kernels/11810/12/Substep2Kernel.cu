#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Substep2Kernel (double *Dens, double *VradInt, double *VthetaInt, double *TemperInt, int nrad, int nsec, double *invdiffRmed, double *invdiffRsup, double *DensInt, int Adiabatic, double *Rmed, double dt, double *VradNew, double *VthetaNew, double *Energy, double *EnergyInt)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dv;

if (i<nrad && j<nsec){

dv = VradInt[(i+1)*nsec + j] - VradInt[i*nsec + j];

if (dv < 0.0)
DensInt[i*nsec + j] = CVNR*CVNR*Dens[i*nsec+j]*dv*dv;
else
DensInt[i*nsec + j] = 0.0;

dv = VthetaInt[i*nsec + (j+1)%nsec] - VthetaInt[i*nsec + j];

if (dv < 0.0)
TemperInt[i*nsec + j] = CVNR*CVNR*Dens[i*nsec+j]*dv*dv;
else
TemperInt[i*nsec + j] = 0.0;
}
}