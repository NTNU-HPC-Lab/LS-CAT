#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void StarThetaKernel (double *Qbase, double *Rmed, int nrad, int nsec, double *dq, double dt)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dxtheta, invdxtheta, dqp, dqm;
if (i<nrad && j<nsec){
if (i<nrad){
dxtheta = 2.0*PI/(double)nsec*Rmed[i];
invdxtheta = 1.0/dxtheta;
}
dqm = (Qbase[i*nsec + j] - Qbase[i*nsec + ((j-1)+nsec)%nsec]);
dqp = (Qbase[i*nsec + (j+1)%nsec] - Qbase[i*nsec + j]);

if (dqp * dqm > 0.0)
dq[i*nsec + j] = dqp*dqm/(dqp+dqm)*invdxtheta;
else
dq[i*nsec + j] = 0.0;
}
}