#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CircumPlanetaryMassKernel (double *Dens, double *Surf, double *CellAbscissa, double *CellOrdinate, double xpl, double ypl, int nrad, int nsec, double HillRadius, double *mdcp0) /* LISTA */
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dist;

if (i<nrad && j<nsec){
dist = sqrt((CellAbscissa[i*nsec + j]-xpl)*(CellAbscissa[i*nsec + j]-xpl) + (CellOrdinate[i*nsec + j]-ypl) * \
(CellOrdinate[i*nsec + j]-ypl));
if (dist < HillRadius) mdcp0[i*nsec + j] = Surf[i]* Dens[i*nsec + j];
else mdcp0[i*nsec + j] = 0.0;
}
}