#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeSpeQtyKernel (double *Label, double *Dens, double *ExtLabel, int nrad, int nsec)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
Label[i*nsec + j] = ExtLabel[i*nsec + j]/Dens[i*nsec + j];
/* Compressive flow if line commentarized
Label[i*nsec + j] = ExtLabel[i*nsec + j] */
}
}