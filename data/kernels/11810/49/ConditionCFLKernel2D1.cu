#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double min2(double a, double b)
{
if (b < a) return b;
return a;
}
__device__ double max2(double a, double b)
{
if (b > a) return b;
return a;
}
__global__ void ConditionCFLKernel2D1 (double *Rsup, double *Rinf, double *Rmed, int nsec, int nrad, double *Vresidual, double *Vtheta, double *Vmoy, int FastTransport, double *SoundSpeed, double *Vrad, double *DT2D)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dxrad, dxtheta, invdt1, invdt2, invdt3, invdt4, dvr, dvt, dt;

if (i > 0 && i<nrad && j<nsec){
dxrad = Rsup[i]-Rinf[i];
dxtheta = Rmed[i]*2.0*PI/(double)nsec;
if (FastTransport) Vresidual[i*nsec + j] = Vtheta[i*nsec + j]-Vmoy[i]; /* Fargo algorithm */
else Vresidual[i*nsec + j] = Vtheta[i*nsec + j];                       /* Standard algorithm */
//Vresidual[i*nsec + nsec] = Vresidual[i*nsec];
invdt1 = SoundSpeed[i*nsec + j]/(min2(dxrad,dxtheta));
invdt2 = fabs(Vrad[i*nsec + j])/dxrad;
invdt3 = fabs(Vresidual[i*nsec + j])/dxtheta;
dvr = Vrad[(i+1)*nsec + j]-Vrad[i*nsec + j];
dvt = Vtheta[i*nsec + (j+1)%nsec]-Vtheta[i*nsec + j];
if (dvr >= 0.0) dvr = 1e-10;
else dvr = -dvr;
if (dvt >= 0.0) dvt = 1e-10;
else dvt = -dvt;
invdt4 = max2(dvr/dxrad, dvt/dxtheta);
invdt4*= 4.0*CVNR*CVNR;
dt = CFLSECURITY/sqrt(invdt1*invdt1+invdt2*invdt2+invdt3*invdt3+invdt4*invdt4);
DT2D[i*nsec + j] = dt; // array nrad*nsec size dt
}
}