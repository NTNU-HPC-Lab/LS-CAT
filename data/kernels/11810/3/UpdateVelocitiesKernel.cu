#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateVelocitiesKernel (double *VthetaInt, double *VradInt, double *invRmed, double *Rmed, double *Rsup, double *Rinf, double *invdiffRmed, double *invdiffRsup, double *Dens, double *invRinf, double *TAURR, double *TAURP, double *TAUPP, double DeltaT, int nrad, int nsec)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

double dphi, invdphi;

/* Now we can update velocities
with the viscous source term
of Navier-Stokes equation */

/* vtheta first */
if (i > 0 && i<nrad-1 && j<nsec){
dphi = 2.0*M_PI/(double)nsec;
invdphi = 1.0/dphi;

VthetaInt[i*nsec +j] += DeltaT*invRmed[i]*((Rsup[i]*TAURP[(i+1)*nsec+ j] - Rinf[i]*TAURP[i*nsec +j])*invdiffRsup[i] + \
(TAUPP[i*nsec +j] - TAUPP[i*nsec + ((j-1)+nsec)%nsec])*invdphi + 0.5*(TAURP[i*nsec + j] + TAURP[(i+1)*nsec +j]))/ \
(0.5*(Dens[i*nsec +j]+Dens[i*nsec + ((j-1)+nsec)%nsec]));
}

/* now vrad */
if (i > 0 && i<nrad && j<nsec){
dphi = 2.0*M_PI/(double)nsec;
invdphi = 1.0/dphi;

VradInt[i*nsec +j] += DeltaT*invRinf[i]*((Rmed[i]*TAURR[i*nsec +j] - Rmed[i-1]*TAURR[(i-1)*nsec + j])*invdiffRmed[i] + \
(TAURP[i*nsec + (j+1)%nsec] - TAURP[i*nsec + j])*invdphi - 0.5*(TAUPP[i*nsec +j] + TAUPP[(i-1)*nsec + j]))/ \
(0.5*(Dens[i*nsec +j] + Dens[(i-1)*nsec + j]));
}
}