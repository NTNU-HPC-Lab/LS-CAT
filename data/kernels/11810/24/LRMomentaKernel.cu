#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LRMomentaKernel (double *RadMomP, double *RadMomM, double *ThetaMomP, double *ThetaMomM, double *Dens, double *Vrad, double *Vtheta, int nrad, int nsec, double *Rmed, double OmegaFrame)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
RadMomP[i*nsec + j] = Dens[i*nsec + j] * Vrad[(i+1)*nsec + j]; // (i+1)*nsec
RadMomM[i*nsec + j] = Dens[i*nsec + j] * Vrad[i*nsec + j];
/* it is the angular momentum -> ThetaMomP */
ThetaMomP[i*nsec + j] = Dens[i*nsec + j] * (Vtheta[i*nsec + (j+1)%nsec]+Rmed[i]*OmegaFrame)*Rmed[i];
ThetaMomM[i*nsec + j] = Dens[i*nsec + j] * (Vtheta[i*nsec + j]+Rmed[i]*OmegaFrame)*Rmed[i];
}
}