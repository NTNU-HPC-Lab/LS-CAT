#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeVelocitiesKernel (double *Vrad, double *Vtheta, double *Dens, double *Rmed, double *ThetaMomP, double *ThetaMomM, double *RadMomP, double *RadMomM, int nrad, int nsec, double OmegaFrame)
{
int j = threadIdx.x + blockDim.x*blockIdx.x;
int i = threadIdx.y + blockDim.y*blockIdx.y;

if (i<nrad && j<nsec){
if (i == 0)
Vrad[i*nsec + j] = 0.0;
else {
Vrad[i*nsec + j] = (RadMomP[(i-1)*nsec + j] + RadMomM[i*nsec + j])/(Dens[i*nsec + j] +
Dens[(i-1)*nsec + j] + 1e-20);
}
Vtheta[i*nsec + j] = (ThetaMomP[i*nsec + ((j-1)+nsec)%nsec] + ThetaMomM[i*nsec + j])/(Dens[i*nsec + j] +
Dens[i*nsec + ((j-1)+nsec)%nsec] + 1e-15)/Rmed[i] - Rmed[i]*OmegaFrame;
/* It was the angular momentum */
}
}