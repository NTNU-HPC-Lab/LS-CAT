#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void kernel_verticalReversal(dev_t *src, dev_t *dst, uint pitch_src, uint pitch_dst, uint pixel_w, uint pixel_h)
{
unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

if (dim_x < pixel_w && dim_y < pixel_h) {
unsigned int rgba = *((uint32_t*)src + dim_y * pitch_src / 4 + dim_x);
*((uint32_t*)dst + (pixel_h - dim_y) * pitch_dst / 4 + dim_x) = rgba;
}
}