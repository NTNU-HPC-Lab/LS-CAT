#include "hip/hip_runtime.h"
#include "includes.h"




__device__ u_char clamp(float t)
{
if (t < 0) {
return 0;
} else if (t > 255){
return 255;
}

return t;
}
__global__ void kernel_colorSpaceRGBAToYUV420P(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h)
{
unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

int r,g,b;
unsigned int rgba;
if (dim_x < w && dim_y < h) {
rgba = *((uint32_t*)dst + dim_y * pitch_dst / 4 + dim_x);
r = (rgba >> 24);
g = (rgba >> 16) & 0xff;
b = (rgba >> 8) & 0xff;
*((u_char*)src + dim_y * pitch_src + dim_x) = clamp(0.299 * r + 0.587 * g + 0.114 * b);
*((u_char*)src + (h + dim_y / 4) * pitch_src + dim_x / 2) = clamp(-0.1687 * r - 0.3313 * g + 0.5 * b + 128);
*((u_char*)src + (h * 5 + dim_y) / 4 * pitch_src + dim_x / 2) = clamp(0.5 * r - 0.4187 * g - 0.0813 * b + 128);
}
}