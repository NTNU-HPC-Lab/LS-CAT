#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void kernel_renderRGBA2Surface(hipSurfaceObject_t surface, dev_t *src, int pitch, int pixel_w, int pixel_h)
{
unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

if (dim_x < pixel_w && dim_y < pixel_h)
{
u_char r,g,b;
r = *((u_char*)src + dim_y * pitch + dim_x * 4);
g = *((u_char*)src + dim_y * pitch + dim_x * 4 + 1);
b = *((u_char*)src + dim_y * pitch + dim_x * 4 + 2);
uchar4 data = make_uchar4(r, g, b, 0xff);
surf2Dwrite(data, surface, dim_x * sizeof(uchar4), dim_y);
}
}