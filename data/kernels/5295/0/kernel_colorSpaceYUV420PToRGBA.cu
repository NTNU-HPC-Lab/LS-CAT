#include "hip/hip_runtime.h"
#include "includes.h"




__device__ u_char clamp(float t)
{
if (t < 0) {
return 0;
} else if (t > 255){
return 255;
}

return t;
}
__global__ void kernel_colorSpaceYUV420PToRGBA(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h)
{
unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

int r,g,b,y,u,v;
if (dim_x < w && dim_y < h) {
y = *((u_char*)src + dim_y * pitch_src + dim_x);
u = *((u_char*)src + (h + dim_y / 4) * pitch_src + dim_x / 2);
v = *((u_char*)src + (h * 5 + dim_y) / 4 * pitch_src + dim_x / 2);
r = clamp(y + 1.402 * (v - 128) + 0.5);
g = clamp(y - 0.34414 * (u - 128) - 0.71414 * (v - 128) + 0.5);
b = clamp(y + 1.772 * (u - 128) + 0.5);
//        *((uint32_t*)dst + dim_y * pitch_dst / 4 + dim_x) = (r << 24) + (g << 16) + (b << 8);
*((u_char*)dst + dim_y * pitch_dst + dim_x * 4) = r;
*((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 1) = g;
*((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 2) = b;
*((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 3) = 255;
}
}