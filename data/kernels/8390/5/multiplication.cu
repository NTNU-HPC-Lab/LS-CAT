#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplication(int * A,int * B,int * C,int N){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if(row<N && col<N){//Si no me fui del arreglo
int sum=0;
for(int i=0;i<N;i++){
sum+=A[row*N+i]*B[i*N+col];
}
C[row*N+col]=sum;
}
}