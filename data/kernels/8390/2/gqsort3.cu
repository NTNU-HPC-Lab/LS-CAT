#include "hip/hip_runtime.h"
#include "includes.h"
#define MAXR(sz) (((sz)+MAXSEQ-1)/MAXSEQ+1)
#define MAXT MAXR(MAXN)
int MAXN;
int MAXSEQ;
int THRN;

//===Definicion de estructuras y funciones utiles===

typedef struct secuence{
int start,end,pivot;
}secuence;

typedef struct block{
secuence seq,parent;
int blockcount,id,bid;
}block;

__global__ void gqsort3(block * blocks,int * d,int * _d){

int id = blockIdx.x,th = threadIdx.x,cth = blockDim.x;
int start = blocks[id].seq.start,end = blocks[id].seq.end;
for(int j=start+th;j<end;j+=cth)
d[j] = _d[j];

return;
}