#include "hip/hip_runtime.h"
#include "includes.h"
#define MAXR(sz) (((sz)+MAXSEQ-1)/MAXSEQ+1)
#define MAXT MAXR(MAXN)
int MAXN;
int MAXSEQ;
int THRN;

//===Definicion de estructuras y funciones utiles===

typedef struct secuence{
int start,end,pivot;
}secuence;

typedef struct block{
secuence seq,parent;
int blockcount,id,bid;
}block;

__global__ void gqsort1(block * blocks,int * d,int * LT,int * GT){

int id = blockIdx.x,th = threadIdx.x,cth = blockDim.x;
int gt=0,lt=0,pivot=blocks[id].seq.pivot;
int start = blocks[id].seq.start,end = blocks[id].seq.end;

if(th==0){
LT[id]=0;
GT[id]=0;
}
__syncthreads();

for(int j=start+th;j<end;j+=cth){
if(d[j]<pivot)lt++;
else if(d[j]>pivot)gt++;
}

atomicAdd(&LT[id],lt);
atomicAdd(&GT[id],gt);

return;
}