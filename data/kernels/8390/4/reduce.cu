#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int * vector,int size,int pot){

int idx = threadIdx.x + blockIdx.x*blockDim.x;
int salto = pot/2;

while(salto){
if(idx<salto && idx+salto<size){
vector[idx]=vector[idx]+vector[idx+salto];
}
__syncthreads();
salto=salto/2;
}

return;
}