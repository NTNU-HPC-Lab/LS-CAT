#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplication(int * A,int * B,int * C,int N,int M,int K){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if(row<N && col<K){//Si no me fui del arreglo
int sum=0;
for(int i=0;i<M;i++){
sum+=A[row*N+i]*B[i*M+col];
}
C[row*N+col]=sum;
}
}