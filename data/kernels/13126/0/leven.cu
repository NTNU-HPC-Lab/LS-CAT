#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void leven(char* a, char* b, char* costs, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i > 0 && i < size) {

costs[0] = i;
int nw = i - 1;
for(int j = 1; j <= size; j++) {
int firstMin = costs[j] < costs[j-1] ? costs[j] : costs[j-1];
// This line is hard to read due to the lack of min() function
int secondMin = 1 + firstMin < a[i - 1] == b[j - 1] ? nw : nw + 1 ? 1 + firstMin : a[i - 1] == b[j - 1] ? nw : nw + 1;
int cj = secondMin;
nw = costs[j];
costs[j] = cj;
}
}

}