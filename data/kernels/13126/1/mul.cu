#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void mul(double* A, double* B, double* C, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;

if(i < size) {
// compute a column
for(int j=0; j < size; j++) {
double sum = 0.0;
for(int k=0; k < size; k++) {
sum += A[ (i*size)+k ] * B[ (k*size)+j ];
}
C[ (i*size)+j ] = sum;
}
// end of column computing
}
}