#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetScaleFactorsKernel(float *indata, float *base, float *stdev, float *factors, int nchans, int processed) {

// NOTE: Filterbank file format coming in
//float mean = indata[threadIdx.x];
float mean = 0.0f;
// NOTE: Depending whether I save STD or VAR at the end of every run
// float estd = stdev[threadIdx.x];
float estd = stdev[threadIdx.x] * stdev[threadIdx.x] * (processed - 1.0f);
float oldmean = base[threadIdx.x];

//float estd = 0.0f;
//float oldmean = 0.0;

float val = 0.0f;
float diff = 0.0;
for (int isamp = 0; isamp < 2 * NACCUMULATE; ++isamp) {
val = indata[isamp * nchans + threadIdx.x];
diff = val - oldmean;
mean = oldmean + diff * factors[processed + isamp + 1];
estd += diff * (val - mean);
oldmean = mean;
}
base[threadIdx.x] = mean;
stdev[threadIdx.x] = sqrtf(estd / (float)(processed + 2 * NACCUMULATE - 1.0f));
// stdev[threadIdx.x] = estd;
}