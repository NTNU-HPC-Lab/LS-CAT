#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgbUtoLab3F_kernel(int width, int height, float gamma, unsigned int* rgbU, float* devL, float* devA, float* devB) {
int x0 = blockDim.x * blockIdx.x + threadIdx.x;
int y0 = blockDim.y * blockIdx.y + threadIdx.y;
if ((x0 < width) && (y0 < height)) {
int index = y0 * width + x0;
unsigned int rgb = rgbU[index];
float r = (float)(rgb & 0xff)/255.0;
float g = (float)((rgb & 0xff00) >> 8)/255.0;
float b = (float)((rgb & 0xff0000) >> 16)/255.0;
r = powf(r, gamma);
g = powf(g, gamma);
b = powf(b, gamma);
float x = (0.412453 * r) +  (0.357580 * g) + (0.180423 * b);
float y = (0.212671 * r) +  (0.715160 * g) + (0.072169 * b);
float z = (0.019334 * r) +  (0.119193 * g) + (0.950227 * b);
/*D65 white point reference */
const float x_ref = 0.950456;
const float y_ref = 1.000000;
const float z_ref = 1.088754;
/* threshold value  */
const float threshold = 0.008856;
x = x / x_ref;
y = y / y_ref;
z = z / z_ref;

float fx =
(x > threshold) ? powf(x,(1.0/3.0)) : (7.787*x + (16.0/116.0));
float fy =
(y > threshold) ? powf(y,(1.0/3.0)) : (7.787*y + (16.0/116.0));
float fz =
(z > threshold) ? powf(z,(1.0/3.0)) : (7.787*z + (16.0/116.0));
/* compute Lab color value */
devL[index] =
(y > threshold) ? (116*powf(y,(1.0/3.0)) - 16) : (903.3*y);
devA[index] = 500.0f * (fx - fy);
devB[index] = 200.0f * (fy - fz);
}
}