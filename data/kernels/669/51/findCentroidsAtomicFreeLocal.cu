#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroidsAtomicFreeLocal(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
int const af_id = blockIdx.x;
int const cluster_id = blockIdx.y;
int const filter_id = threadIdx.x;
int* filter_responses = &responses[filter_id*nPixels];

int local_responses = 0;
int local_count = 0;

int pixel_start = af_id*afLocal;
int pixel_end = (af_id+1)*afLocal;

pixel_end = pixel_end>nPixels?nPixels:pixel_end;

for (int i=pixel_start; i<pixel_end; i++)
{
if (cluster[i] == cluster_id)
{
local_responses += filter_responses[i];
local_count++;
}
}

int idx = af_id * gridDim.y*blockDim.x + filter_id*32 + cluster_id;
centroidMass[idx] = local_responses;
centroidCount[idx] = local_count;
}