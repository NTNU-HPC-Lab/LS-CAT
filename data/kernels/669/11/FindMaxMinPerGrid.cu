#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMaxMinPerGrid(int p_nGridSize, int p_nEigNum, float* p_devMax, float* p_devMin, float* p_devReduceMax, float* p_devReduceMin, int p_nMaxLevel)
{
__shared__ float MaxReduce[XBLOCK*(MAXEIGNUM - 1)];
__shared__ float MinReduce[XBLOCK*(MAXEIGNUM - 1)];

int taskPerTh = (p_nGridSize + XBLOCK - 1)/XBLOCK;
// First Assignment

if (threadIdx.x < p_nGridSize)
{
for (int i = 0; i < p_nEigNum - 1; i++)
{
MaxReduce[i*XBLOCK + threadIdx.x] = p_devMax[threadIdx.x + i * p_nGridSize];
MinReduce[i*XBLOCK + threadIdx.x] = p_devMin[threadIdx.x + i * p_nGridSize];
}
}

// First Reduction
for (int i = 1; i < taskPerTh; i++)
{
int curIndex = threadIdx.x + i * XBLOCK;
if (curIndex < p_nGridSize)
{
for (int j = 0; j < p_nEigNum - 1; j++)
{
if (MaxReduce[j*XBLOCK + threadIdx.x] < p_devMax[curIndex + j * p_nGridSize])
{
MaxReduce[j*XBLOCK + threadIdx.x] = p_devMax[curIndex + j * p_nGridSize];
}
if (MinReduce[j*XBLOCK + threadIdx.x] > p_devMin[curIndex + j * p_nGridSize])
{
MinReduce[j*XBLOCK + threadIdx.x] = p_devMin[curIndex + j * p_nGridSize];
}
}
}
}
__syncthreads();

//The Reductions Thereafter
int mask = 1;
for (int level = 0; level < p_nMaxLevel; level++)
{
if ((threadIdx.x & mask) == 0)
{
int index1 = threadIdx.x;
int index2 = (1 << level) + threadIdx.x;
if (index2 < p_nGridSize)
{
for (int i = 0; i < p_nEigNum - 1; i++)
{
if (MaxReduce[i*XBLOCK + index1] < MaxReduce[i*XBLOCK + index2])
{
MaxReduce[i*XBLOCK + index1] = MaxReduce[i*XBLOCK + index2];
}
if (MinReduce[i*XBLOCK + index1] > MinReduce[i*XBLOCK + index2])
{
MinReduce[i*XBLOCK + index1] = MinReduce[i*XBLOCK + index2];
}
}
}
}
__syncthreads();
mask = (mask<<1)|1;
}

//Write max and min into global memory
if (threadIdx.x == 0)
{
for (int i = 0; i < p_nEigNum - 1; i++)
{
p_devReduceMax[i] = MaxReduce[i*XBLOCK];
p_devReduceMin[i] = MinReduce[i*XBLOCK];
}
}

}