#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
__shared__ int localMasses[64*17];
__shared__ unsigned int localCounts[64];
int pixel = blockDim.x * blockIdx.x + threadIdx.x;
if (threadIdx.x < 64) {
for (int i = 0; i < 17; i++) {
localMasses[64 * i + threadIdx.x] = 0;
}
localCounts[threadIdx.x] = 0;
}
__syncthreads();
if (pixel < nPixels) {
int myCluster = cluster[pixel];
int myIndex = pixel;
for(int filter = 0; filter < 17; filter++) {
int myElement = responses[myIndex];
atomicAdd(localMasses + filter * 64 + myCluster, myElement);
myIndex += nPixels;
}
}
__syncthreads();
if (threadIdx.x < 64) {
for (int filter = 0; filter < 17; filter++) {
atomicAdd(centroidMass + filter * 64 + threadIdx.x, localMasses[threadIdx.x + filter * 64]);
localMasses[threadIdx.x + filter * 64] = 0;
}
}
__syncthreads();
if (pixel < nPixels) {
int myCluster = cluster[pixel];
// yunsup fixed
int myIndex = pixel + nPixels*17;
for(int filter = 17; filter < 34; filter++) {
int myElement = responses[myIndex];
atomicAdd(localMasses + filter * 64 + myCluster, myElement);
myIndex += nPixels;
}
atomicInc(localCounts + myCluster, 100000000);
}
__syncthreads();
if (threadIdx.x < 64) {
for (int filter = 17; filter < 34; filter++) {
atomicAdd(centroidMass + filter * 64 + threadIdx.x, localMasses[threadIdx.x + filter * 64]);
localMasses[threadIdx.x + filter * 64] = 0;
}
atomicAdd(centroidCount + threadIdx.x, localCounts[threadIdx.x]);
}
}