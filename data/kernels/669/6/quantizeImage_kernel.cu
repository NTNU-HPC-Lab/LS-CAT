#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void quantizeImage_kernel(uint width, uint height, uint nbins, float* devInput, int* devOutput) {
int x0 = blockDim.x * blockIdx.x + threadIdx.x;
int y0 = blockDim.y * blockIdx.y + threadIdx.y;
if ((x0 < width) && (y0 < height)) {
int index = y0 * width + x0;
float input = devInput[index];
int output = (int)floorf(input * (float)nbins);
if (output == nbins) {
output = nbins - 1;
}
devOutput[index] = output;
}
}