#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unMirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, float* devInput, float* devOutput) {
int x0 = blockDim.x * blockIdx.x + threadIdx.x;
int y0 = blockDim.y * blockIdx.y + threadIdx.y;
if ((x0 < borderWidth) && (y0 < borderHeight)) {
int x1 = x0 + border;
int y1 = y0 + border;
devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
}
}