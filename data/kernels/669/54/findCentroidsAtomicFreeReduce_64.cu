#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroidsAtomicFreeReduce_64(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
int const af_id = blockIdx.x;
int const cluster_id = blockIdx.y;
int const filter_id = threadIdx.x;

int local_mass = 0;
int local_count = 0;

if (af_id == 0)
{
int idx0 = filter_id*64 + cluster_id;

for (int i=0; i<gridDim.x; i++)
{
int idxother = i * gridDim.y*blockDim.x + idx0;

local_mass += centroidMass[idxother];
local_count += centroidCount[idxother];
}

centroidMass[idx0] = local_mass;
centroidCount[idx0] = local_count;
}
}