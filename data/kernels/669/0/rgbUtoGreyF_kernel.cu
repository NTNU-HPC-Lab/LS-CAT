#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgbUtoGreyF_kernel(int width, int height, unsigned int* rgbU, float* grey) {
int x = blockDim.x * blockIdx.x + threadIdx.x;
int y = blockDim.y * blockIdx.y + threadIdx.y;
if ((x < width) && (y < height)) {
int index = y * width + x;
unsigned int rgb = rgbU[index];
float r = (float)(rgb & 0xff)/255.0;
float g = (float)((rgb & 0xff00) >> 8)/255.0;
float b = (float)((rgb & 0xff0000) >> 16)/255.0;
grey[index] =
(0.29894 * r)
+ (0.58704 * g)
+ (0.11402 * b);
}
}