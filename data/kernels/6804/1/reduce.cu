#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(double *a,double *z, int sizeOut){
int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid > N/2)return;

extern __shared__ double subTotals[];
subTotals[threadIdx.x]=(a[tid*2]+a[tid*2+1])/2;//sum every two values using all threads
__syncthreads();
int level=2;
while ((blockDim.x/level) >= sizeOut){//keep halving values until sizeout remains
if(threadIdx.x % level==0){//use half threads every iteration
subTotals[threadIdx.x]=(subTotals[threadIdx.x]+subTotals[threadIdx.x+(level/2)])/2;
}
__syncthreads();//we have to sync threads every time here :(
level = level * 2;
}
level = level /2;
if(threadIdx.x % level==0){
z[tid/level] = subTotals[threadIdx.x];
}
}