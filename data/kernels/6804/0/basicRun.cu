#include "hip/hip_runtime.h"
#include "includes.h"
//CUDA reduction algorithm. simple approach
//Tom Dale
//11-20-18


using namespace std;
#define N 100000//number of input values
#define R 100//reduction factor
#define F (1+((N-1)/R))//how many values will be in the final output


//basicRun will F number of threads go through R number of values and put the average in z[tid]




__global__ void basicRun(double *a,double *z){
int tid = blockDim.x*blockIdx.x + threadIdx.x;
if(tid > F) return;
double avg=0;
for(int i= 0;i<R;i++){//get sum of input values in this threads domain
avg += a[i+tid*R];
}
z[tid]=avg/R;//divide sum by total number of input values to get average
}