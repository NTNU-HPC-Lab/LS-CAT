#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyVectors(float* A, float* B, float*C,int WIDTH,int HEIGHT) {
int x = threadIdx.x + blockIdx.x*blockDim.x;
int y = threadIdx.y + blockIdx.y*blockDim.y;

if (x<WIDTH && y<HEIGHT) {

double result = 0.0;

for (int i=0;i<WIDTH;i++)
result+=A[y*WIDTH+i]*B[i*WIDTH+x];

C[y*WIDTH+x] = result;
}
}