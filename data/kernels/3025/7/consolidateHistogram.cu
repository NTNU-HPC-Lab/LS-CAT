#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void consolidateHistogram(ulong*blockHistograms,ulong* cudaHistogram,uint numBlocks) {
int tid = threadIdx.x;

for (uint j=0;j<numBlocks;j++) {
cudaHistogram[tid]+=blockHistograms[j*256+tid];
}
}