#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N){
const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
const int threadN = gridDim.x * blockDim.x;
float sum = 0;
for(int pos = tid; pos < N; pos += threadN)
sum += d_Input[pos];

d_Result[tid] = sum;
}