#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float *xs, const float *ys, float *out, const unsigned int N)
{
unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

if (idx < N)
out[idx] = xs[idx] + ys[idx];
}