#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = id; i < valid_voxel_num; i += stride) {
e_x_cov_x[i] *= gauss_d2;
}
}