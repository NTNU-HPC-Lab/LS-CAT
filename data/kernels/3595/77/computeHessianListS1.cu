#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeHessianListS1(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double gauss_d1, double gauss_d2, double *hessians, double *e_x_cov_x, double *tmp_hessian, double *cov_dxd_pi, double *point_gradients, int valid_voxel_num)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int row = blockIdx.y;
int col = blockIdx.z;

if (row < 6 && col < 6) {
double *cov_dxd_pi_mat0 = cov_dxd_pi + row * valid_voxel_num;
double *cov_dxd_pi_mat1 = cov_dxd_pi_mat0 + 6 * valid_voxel_num;
double *cov_dxd_pi_mat2 = cov_dxd_pi_mat1 + 6 * valid_voxel_num;
double *tmp_h = tmp_hessian + col * valid_voxel_num;
double *h = hessians + (row * 6 + col) * valid_points_num;
double *tmp_pg0 = point_gradients + col * valid_points_num;
double *tmp_pg1 = tmp_pg0 + 6 * valid_points_num;
double *tmp_pg2 = tmp_pg1 + 6 * valid_points_num;

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];
double d_x = static_cast<double>(trans_x[pid]);
double d_y = static_cast<double>(trans_y[pid]);
double d_z = static_cast<double>(trans_z[pid]);

double pg0 = tmp_pg0[i];
double pg1 = tmp_pg1[i];
double pg2 = tmp_pg2[i];

double final_hessian = 0.0;

for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
//Transformed coordinates
int vid = voxel_id[j];

double tmp_ex = e_x_cov_x[j];

if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
double cov_dxd0 = cov_dxd_pi_mat0[j];
double cov_dxd1 = cov_dxd_pi_mat1[j];
double cov_dxd2 = cov_dxd_pi_mat2[j];

tmp_ex *= gauss_d1;

final_hessian += -gauss_d2 * ((d_x - centroid_x[vid]) * cov_dxd0 + (d_y - centroid_y[vid]) * cov_dxd1 + (d_z - centroid_z[vid]) * cov_dxd2) * tmp_h[j] * tmp_ex;
final_hessian += (pg0 * cov_dxd0 + pg1 * cov_dxd1 + pg2 * cov_dxd2) * tmp_ex;
}
}

h[i] = final_hessian;
}
}
}