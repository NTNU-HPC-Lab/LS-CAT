#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

float CValue = 0;

int Row = blockIdx.y*16 + threadIdx.y;
int Col = blockIdx.x*16 + threadIdx.x;

for (int k = 0; k < (16 + ACols - 1)/16; k++) {

for (int n = 0; n < 16; ++n)
if ((k*16 + n < ACols && Row < ARows) && (k*16 + n < BRows && Col < BCols))
CValue += A[Row*ACols + k*16 + n] * B[(k*16 + n)*BCols + Col];

}

if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}