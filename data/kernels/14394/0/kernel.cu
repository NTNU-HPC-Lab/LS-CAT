#include "hip/hip_runtime.h"
#include "includes.h"
// CUDA runtime

// helper functions and utilities to work with CUDA


extern "C"
__global__ void kernel(int* data, int size)
{
int id = blockDim.x * blockIdx.x + threadIdx.x;
data[id] = id;
}