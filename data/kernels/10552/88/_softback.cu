#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _softback(int nrows, int ncols, float *y, float *dy) {
/* y is layer output, i.e. unnormalized log probabilities.
On output y will contain normalized probabilities.
Conceptually this is a forward calculation but we do it here for efficiency.
dy is the label matrix: each column is a one-hot vector indicating the correct label.
On output dy will be the gradient of softmax loss wrt probabilities.
*/
float y0, sum;
int i0, i1;
int col = threadIdx.x + blockIdx.x * blockDim.x;
while (col < ncols) {
i0 = col * nrows;
i1 = i0  + nrows;
y0 = -INFINITY;
//y0 = y[i0];
for (int i=i0; i<i1; i++) {
if (y[i] > y0) {
y0 = y[i];
}
}
sum = 0;
for (int i=i0; i<i1; i++) {
y[i] = exp(y[i]-y0);
sum += y[i];
}
for (int i=i0; i<i1; i++) {
y[i] /= sum;
dy[i] = (y[i] - dy[i]) / ncols;
}
col += blockDim.x * gridDim.x;
}
}