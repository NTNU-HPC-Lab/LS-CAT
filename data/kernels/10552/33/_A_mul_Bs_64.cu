#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) {
int s0, s1, sp, sc, sr, x0, xr, k0, k1, kp;
double sv, xv;
sc = threadIdx.x + blockIdx.x * blockDim.x;
while (sc < ns) {	// sc: 0-based column for s and k to be processed
k0 = mx*sc;		// k[k0]: first element of k[:,sc]
k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
for (kp = k0; kp < k1; kp++) k[kp] = 0;
s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
for (sp = s0; sp < s1; sp++) {
sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
sv = sval[sp];	// sv: s[sr,sc] (0-based), this value multiplies the sr'th column of x
x0 = mx*sr;	// x[x0]: first element of x[:,sr]
for (xr = 0; xr < mx; xr++) {
xv = x[x0+xr];     // xv: x[xr,sr], now we can set k[xr,sc]
k[k0+xr] += xv*sv;
}
}
sc += blockDim.x * gridDim.x;
}
}