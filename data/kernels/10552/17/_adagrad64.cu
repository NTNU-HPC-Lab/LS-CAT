#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _adagrad64(int n, double eps, double *dw2, double *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
dw2[i] += dw[i] * dw[i];
dw[i] /= sqrt(dw2[i] + eps);
i += blockDim.x * gridDim.x;
}
}