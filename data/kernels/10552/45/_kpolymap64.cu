#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kpolymap64(int n, double *k, double c, double d) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
k[i] = pow(k[i] + c, d);
i += blockDim.x * gridDim.x;
}
}