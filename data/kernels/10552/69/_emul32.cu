#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _emul32(int n, float *x, float *y) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
y[i] *= x[i];
i += blockDim.x * gridDim.x;
}
}