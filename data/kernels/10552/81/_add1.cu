#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _add1(int n, float val, float *x) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
x[i] += val;
i += blockDim.x * gridDim.x;
}
}