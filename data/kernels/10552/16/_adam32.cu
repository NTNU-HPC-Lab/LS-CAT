#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _adam32(int n, int t, double eps, double b1, double b2, float *fstm, float *scndm, float *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
fstm[i] = b1*fstm[i] + (1-b1)*dw[i];
scndm[i] = b2*scndm[i] + (1-b2)*(dw[i] *dw[i]);
dw[i] = (fstm[i] / (1 - pow(b1,(double)t))) / (sqrt(scndm[i] / (1 - pow(b2,(double)t))) + eps);

i += blockDim.x * gridDim.x;
}
}