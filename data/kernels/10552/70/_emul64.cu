#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _emul64(int n, double *x, double *y) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
y[i] *= x[i];
i += blockDim.x * gridDim.x;
}
}