#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _l1reg64(int n, double l1, double *w, double *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (w[i] > 0) dw[i] += l1;
else if (w[i] < 0) dw[i] -= l1;
i += blockDim.x * gridDim.x;
}
}