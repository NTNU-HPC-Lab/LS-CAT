#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss32sum(int xrows, int xcols, float *x, float *xx) {
int i, j, x0, x1;
double sum;
j = threadIdx.x + blockIdx.x * blockDim.x;
while (j < xcols) {
x0 = j*xrows; x1 = x0+xrows;
sum = 0;
for (i=x0; i<x1; i++) sum += x[i]*x[i];
xx[j] = sum;
j += blockDim.x * gridDim.x;
}
}