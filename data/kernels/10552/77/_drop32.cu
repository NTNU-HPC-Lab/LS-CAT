#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _drop32(int n, float *x, float *y, float *xmask, double dropout, double scale) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (xmask[i] < dropout) y[i] = 0;
else y[i] = x[i] * scale;
i += blockDim.x * gridDim.x;
}
}