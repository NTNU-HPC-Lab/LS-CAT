#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _drop64(int n, double *x, double *y, double *xmask, double dropout, double scale) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (xmask[i] < dropout) y[i] = 0;
else y[i] = x[i] * scale;
i += blockDim.x * gridDim.x;
}
}