#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _reluforw(int n, float *y) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (y[i] < 0) y[i] = 0;
i += blockDim.x * gridDim.x;
}
}