#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _adagrad(int n, float eps, float *dw2, float *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
dw2[i] += dw[i] * dw[i];
dw[i] /= (eps + sqrt(dw2[i]));
i += blockDim.x * gridDim.x;
}
}