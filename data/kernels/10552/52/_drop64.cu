#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _drop64(int n, double *x, double *xmask, double dropout, double scale) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (xmask[i] < dropout) x[i] = 0;
else x[i] *= scale;
i += blockDim.x * gridDim.x;
}
}