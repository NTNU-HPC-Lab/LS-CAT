#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float g, float *k) {
// assume x(mx,nd) and s(nd,ns) are in 1-based csc format
// assume k(mx,ns) has been allocated and zeroed out
int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
float sv, xv, xs;
sc = threadIdx.x + blockIdx.x * blockDim.x;
k0 = mx*sc;		// k[k0]: first element of k[:,sc]
k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
while (sc < ns) {	// sc: 0-based column for s
s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
for (sp = s0; sp < s1; sp++) {
sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
sv = sval[sp];	// sv: s[sr,sc] (0-based)
xc = sr;		// xc: 0-based column for x (=sr)
x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
for (xp = x0; xp < x1; xp++) {
xr = xrow[xp]-1; // xr: 0-based row for x
xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
xs = xv - sv;
k[k0+xr] += xs*xs; // k += (xi-si)^2
}
}
for (kp = k0; kp < k1; kp++) {
k[kp] = exp(-g*k[kp]); // k = exp(-g*sum((xi-si)^2))
}
sc += blockDim.x * gridDim.x;
}
}