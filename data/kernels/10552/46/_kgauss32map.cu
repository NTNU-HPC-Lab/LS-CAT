#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss32map(int nx, int ns, float *x2, float *s2, float *k, float g) {
int i, n, xi, si;
i = threadIdx.x + blockIdx.x * blockDim.x;
n = nx*ns;
while (i < n) {
xi = (i % nx);
si = (i / nx);
k[i] = exp(-g * (x2[xi] + s2[si] - 2*k[i]));
i += blockDim.x * gridDim.x;
}
}