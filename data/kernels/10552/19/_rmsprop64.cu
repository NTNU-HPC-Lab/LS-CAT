#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _rmsprop64(int n, double eps, double rho, double *dw2, double *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
dw2[i] = dw2[i] * rho + (1 - rho) * dw[i] * dw[i];
dw[i] /= sqrt(dw2[i] + eps);
i += blockDim.x * gridDim.x;
}
}