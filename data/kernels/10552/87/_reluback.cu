#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _reluback(int n, float *y, float *dy) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (y[i] <= 0) dy[i] = 0;
i += blockDim.x * gridDim.x;
}
}