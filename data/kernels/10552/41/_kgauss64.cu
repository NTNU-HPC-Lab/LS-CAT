#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double g) {
int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
double d, dd;
i = threadIdx.x + blockIdx.x * blockDim.x;
n = nx*ns;
while (i < n) {
xc = i % nx;
sc = i / nx;
x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
s1 = scol[sc]-1; s2 = scol[sc+1]-1;
dd = 0;
while ((x1 < x2) || (s1 < s2)) {
xr = ((x1 < x2) ? xrow[x1] : INT_MAX);
sr = ((s1 < s2) ? srow[s1] : INT_MAX);
d = ((sr < xr) ? sval[s1++] :
(xr < sr) ? xval[x1++] :
(xval[x1++]-sval[s1++]));
dd += d*d;
}
k[i] = exp(-g * dd);
i += blockDim.x * gridDim.x;
}
}