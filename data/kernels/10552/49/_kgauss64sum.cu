#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss64sum(int xrows, int xcols, double *x, double *xx) {
int i, j, x0, x1;
double sum;
j = threadIdx.x + blockIdx.x * blockDim.x;
while (j < xcols) {
x0 = j*xrows; x1 = x0+xrows;
sum = 0;
for (i=x0; i<x1; i++) sum += x[i]*x[i];
xx[j] = sum;
j += blockDim.x * gridDim.x;
}
}