#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
i = threadIdx.x + blockIdx.x * blockDim.x;
n = nx*ns;
while (i < n) {
double ki = 0;
xc = i % nx;
sc = i / nx;
x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
s1 = scol[sc]-1; s2 = scol[sc+1]-1;
while ((x1 < x2) && (s1 < s2)) {
xr = xrow[x1]; sr = srow[s1];
if (sr < xr) s1++;
else if (xr < sr) x1++;
else ki += xval[x1++]*sval[s1++];
}
k[i] = ki;
i += blockDim.x * gridDim.x;
}
}