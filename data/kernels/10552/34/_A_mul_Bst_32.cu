#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _A_mul_Bst_32(int my, int xc, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
// dw[wr,wc] += dy[yr,yc] * x[xr,xc]  where wr=yr, wc=xr, yc=xc
int t, n, xp, xr, yp, yr, wp;
t = threadIdx.x + blockIdx.x * blockDim.x;
n = xcol[xc+1] - xcol[xc];
while (t < n) {
xp = xcol[xc] + t - 1;
xr = xrow[xp] - 1;
for (yr = 0; yr < my; yr++) {
yp = yr + xc * my;
wp = yr + xr * my;
dw[wp] += dy[yp] * xval[xp];
}
t += blockDim.x * gridDim.x;
}
}