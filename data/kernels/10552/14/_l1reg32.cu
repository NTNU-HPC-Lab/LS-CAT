#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _l1reg32(int n, double l1, float *w, float *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (w[i] > 0) dw[i] += l1;
else if (w[i] < 0) dw[i] -= l1;
i += blockDim.x * gridDim.x;
}
}