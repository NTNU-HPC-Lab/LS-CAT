#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _rmsprop32(int n, double eps, double rho, float *dw2, float *dw) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
dw2[i] = dw2[i] * rho + (1 - rho) * dw[i] * dw[i];
dw[i] /= sqrt(dw2[i] + eps);
i += blockDim.x * gridDim.x;
}
}