#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
double sv, xv;
sc = threadIdx.x + blockIdx.x * blockDim.x;
while (sc < ns) {	// sc: 0-based column for s
k0 = mx*sc;		// k[k0]: first element of k[:,sc]
k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
for (kp = k0; kp < k1; kp++) k[kp] = 0;
s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
for (sp = s0; sp < s1; sp++) {
sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
sv = sval[sp];	// sv: s[sr,sc] (0-based)
xc = sr;		// xc: 0-based column for x (=sr)
x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
for (xp = x0; xp < x1; xp++) {
xr = xrow[xp]-1; // xr: 0-based row for x
xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
k[k0+xr] += xv*sv;
}
}
sc += blockDim.x * gridDim.x;
}
}