#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss64d(int nx, int ns, int nd, double *x, double *s, double *k, double g) {
int i, j, n, xj, sj;
double d, dd;
i = threadIdx.x + blockIdx.x * blockDim.x;
n = nx*ns;
while (i < n) {
xj = (i % nx)*nd;
sj = (i / nx)*nd;
dd = 0;
for (j = 0; j < nd; j++) {
d = x[xj++]-s[sj++];
dd += d*d;
}
k[i] = exp(-g * dd);
i += blockDim.x * gridDim.x;
}
}