#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _logploss(int nrows, int ncols, float *y, float *dy) {
/* Similar to softmaxloss, except y is assumed normalized logp and is not overwritten.
y is layer output, i.e. normalized log probabilities.
dy is the label matrix: each column is a one-hot vector indicating the correct label.
On output dy will be the gradient of softmax loss wrt log probabilities.
*/
int col = threadIdx.x + blockIdx.x * blockDim.x;
int i0, i1;
while (col < ncols) {
i0 = col * nrows;
i1 = i0  + nrows;
for (int i=i0; i<i1; i++) {
dy[i] = (exp(y[i]) - dy[i]) / ncols;
}
col += blockDim.x * gridDim.x;
}
}