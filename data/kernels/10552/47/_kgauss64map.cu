#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss64map(int nx, int ns, double *x2, double *s2, double *k, double g) {
int i, n, xi, si;
i = threadIdx.x + blockIdx.x * blockDim.x;
n = nx*ns;
while (i < n) {
xi = (i % nx);
si = (i / nx);
k[i] = exp(-g * (x2[xi] + s2[si] - 2*k[i]));
i += blockDim.x * gridDim.x;
}
}