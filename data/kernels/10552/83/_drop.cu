#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _drop(int n, float *x, float *xmask, float dropout, float scale) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
while (i < n) {
if (xmask[i] < dropout) x[i] = 0;
else x[i] *= scale;
i += blockDim.x * gridDim.x;
}
}