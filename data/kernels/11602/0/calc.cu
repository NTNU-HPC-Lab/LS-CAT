#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef INFINITY
/* INFINITY is supported */
#endif

float **A, **D, *d2; //Table A distance, D minimum distance,d2 tempTable 1-d

__global__ void calc(float *d_D, int n, int k){
int i = blockIdx.x * blockDim.x + threadIdx.x;   //We find i & j in the Grid of threads
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (d_D[i + j*n] > d_D[i + k*n] + d_D[k + j*n]) d_D[i + j*n] = d_D[i + k*n] + d_D[k + j*n];  //Every thread calculates its proper value
}