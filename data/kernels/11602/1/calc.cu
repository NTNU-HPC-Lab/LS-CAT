#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc(float *d_D, int n, int k){ //kernel
__shared__ float s_d[3*256]; //shared in block table of floats (size 3*number threads/block)
int i = blockIdx.x * blockDim.x + threadIdx.x;  //We find i & j in the Grid of threads
int j = blockIdx.y * blockDim.y + threadIdx.y;
int b_index = 3 * (threadIdx.x + blockDim.x*threadIdx.y); //Calculation of initial index in shared table s_d
s_d[b_index] = d_D[i + j*n];  //Pass values from device table to shared
s_d[b_index + 1] = d_D[i + k*n];
s_d[b_index + 2] = d_D[k + j*n];
if (s_d[b_index] > s_d[b_index + 1] + s_d[b_index + 2]) s_d[b_index] = s_d[b_index + 1] + s_d[b_index + 2]; //Calculation of new distance value
d_D[i + j*n] = s_d[b_index]; //Pass the values back to the table s_d
}