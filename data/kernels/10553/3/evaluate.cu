#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void evaluate(chromosome *cudaChromo,int *devValue,int *devWeight, int numele)
{
int idx = threadIdx.x+blockDim.x*blockIdx.x;
for (int i = 0; i < numele; i++){
if (cudaChromo[idx].chromo[i])
cudaChromo[idx].value += devValue[i];
cudaChromo[idx].weight += (cudaChromo[idx].chromo[i] ? 1 : 0)*devWeight[i];
}

}