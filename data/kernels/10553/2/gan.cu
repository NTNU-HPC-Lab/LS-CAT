#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void gan(chromosome *cudaChromo, chromosome* cudaNewpopulation,const int capacity)
{
int idx = threadIdx.x + blockIdx.x*blockDim.x;
if (idx < N) {
for (int i = idx; i < NewN;i+=N)
if (cudaNewpopulation[i].weight<=capacity&&cudaNewpopulation[i].value>cudaChromo[idx].value)
cudaChromo[idx] = cudaNewpopulation[i];
}
}