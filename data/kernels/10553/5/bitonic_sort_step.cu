#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void bitonic_sort_step(chromosome *cudanewpopulation, int j, int k)
{
unsigned int i, ixj; /* Sorting partners: i and ixj */
i = threadIdx.x + blockDim.x * blockIdx.x;
ixj = i^j;
printf("                    %d                        \n", i);

/* The threads with the lowest ids sort the array. */
if ((ixj) > i) {
if ((i&k) != 0) {
/* Sort ascending */
if (cudanewpopulation[i].value < cudanewpopulation[ixj].value) {
/* exchange(i,ixj); */
chromosome temp = cudanewpopulation[i];
cudanewpopulation[i] = cudanewpopulation[ixj];
cudanewpopulation[ixj] = temp;

}
}
}

}