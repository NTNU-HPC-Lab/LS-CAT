#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void initOne(chromosome *cudaChromo, const int numele,int *devValue,int *devWeight)
{
if (blockIdx.x < N){
int idx = threadIdx.x + blockIdx.x*blockDim.x;
idx %= numele;
if (blockIdx.x == idx)
{
cudaChromo[blockIdx.x].chromo[idx] = true;
cudaChromo[blockIdx.x].value = devValue[idx];
cudaChromo[blockIdx.x].weight = devValue[idx];
}
else
cudaChromo[blockIdx.x].chromo[idx] = false;
}
}