#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void initpopulation(chromosome *cudaChromo,int seed,const int numofeles,int *devValue,int* devWeight)
{
if (blockIdx.x < N){
int idx = (threadIdx.x + blockIdx.x*blockDim.x);
hiprandState state;
hiprand_init(seed, idx, 1, &state);
idx %= numofeles;
bool tmp = hiprand(&state) % 2 == 1 ? true : false;
cudaChromo[blockIdx.x].chromo[idx] = tmp;
}
}