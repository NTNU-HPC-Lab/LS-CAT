#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
int f1 = index / n;
int f2 = index % n;
if (f2 <= f1) return;

float sum = 0;
float norm1 = 0;
float norm2 = 0;
int b, i;
for(b = 0; b <  batch; ++b){
for(i = 0; i < size; ++i){
int i1 = b * size * n + f1 * size + i;
int i2 = b * size * n + f2 * size + i;
sum += output[i1] * output[i2];
norm1 += output[i1] * output[i1];
norm2 += output[i2] * output[i2];
}
}
norm1 = sqrt(norm1);
norm2 = sqrt(norm2);
float norm = norm1 * norm2;
sum = sum / norm;
for(b = 0; b <  batch; ++b){
for(i = 0; i < size; ++i){
int i1 = b * size * n + f1 * size + i;
int i2 = b * size * n + f2 * size + i;
delta[i1] += - scale * sum * output[i2] / norm;
delta[i2] += - scale * sum * output[i1] / norm;
}
}
}