#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
for(i = 0; i < n; ++i){
int val = input[i*stride];
largest = (val>largest) ? val : largest;
}
for(i = 0; i < n; ++i){
float e = expf(input[i*stride]/temp - largest/temp);
sum += e;
output[i*stride] = e;
}
for(i = 0; i < n; ++i){
output[i*stride] /= sum;
}
}
__device__ void softmax_device(int n, float *input, float temp, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
for(i = 0; i < n; ++i){
int val = input[i];
largest = (val>largest) ? val : largest;
}
for(i = 0; i < n; ++i){
float e = exp(input[i]/temp - largest/temp);
sum += e;
output[i] = e;
}
for(i = 0; i < n; ++i){
output[i] /= sum;
}
}
__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= batch*groups) return;
int b = id / groups;
int g = id % groups;
softmax_device(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}