#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initMemory(size_t position, size_t* array)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
array[position + idx] = idx;
}