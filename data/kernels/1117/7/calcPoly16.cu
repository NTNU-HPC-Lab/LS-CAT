#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__device__ void calculateZ(int* result, int a0 , int a1 , int a2 , int a3 , int a4 , int a5 , int a6 , int a7 , int a8 , int a9 , int a10, int a11, int a12, int a13, int a14, int a15, int x, int modulus )
{
*result = (
a0 + x*(
a1 + x*(
a2 + x*(
a3 + x*(
a4 + x*(
a5 + x*(
a6 + x*(
a7 + x*(
a8 + x*(
a9 + x*(
a10 + x*(
a11 + x*(
a12 + x*(
a13 + x*(
a14 + x*(
a15 + x*(1) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus
) % modulus;
}
__global__ void calcPoly16()
{
int f_base = threadIdx.z; // is 0-4
int a1  = threadIdx.y;
int a2  = threadIdx.x;

int a3  =  blockIdx.z       & 0xF;
int a4  = (blockIdx.z >> 4) & 0xF;
int a5  = (blockIdx.z >> 8) & 0xF;

int a6  =  blockIdx.y       & 0xF;
int a7  = (blockIdx.y >> 4) & 0xF;
int a8  = (blockIdx.y >> 8) & 0xF;

int a9  =   blockIdx.x        & 0xF;
int a10 = (blockIdx.x >> 4)  & 0xF;
int a11 = (blockIdx.x >> 8)  & 0xF;
int a12 = (blockIdx.x >> 12) & 0xF;
int a13 = (blockIdx.x >> 16) & 0xF;
int a14 = (blockIdx.x >> 20) & 0xF;
int a15 = (blockIdx.x >> 24) & 0xF;

int MOD = 17; // Compiler seems automatically optimized % 16 to & 0xF

int a0 = 4*f_base;

// ??? Does this array the same across all threads??
int Y[10] = {0};
// Make sure distinct values
for(int x = 0; x < 8; x++)
{
int result = 0;
calculateZ(&result,
a0, a1, a2, a3,
a4, a5, a6, a7,
a8, a9, a10, a11,
a12, a13, a14, a15,
x, MOD);
int result0 = (result + 0) % MOD;
int result1 = (result + 1) % MOD;
int result2 = (result + 2) % MOD;
int result3 = (result + 3) % MOD;

Y[0] |= (1 << result0);
Y[1] |= (1 << result1);
Y[2] |= (1 << result2);
Y[3] |= (1 << result3);
}

for (int idx_fg=0;idx_fg<4;idx_fg++)
{
//                 FEDCBA9876543210
if (Y[idx_fg] == 0b0011001101010011)
{
// Let's check it whether in back row
for(int x = 8; x < 16; x++)
{
int result = 0;
calculateZ(&result,
a0+idx_fg, a1, a2, a3,
a4, a5, a6, a7,
a8, a9, a10, a11,
a12, a13, a14, a15,
x, MOD);
int result_b = (result) % MOD;

Y[4+idx_fg] |= (1 << result_b);
}

//                    FEDCBA9876543210
if ( Y[4+idx_fg] == 0b1100110010101100)
{
int res[16];
for(int tmpi = 0; tmpi<16;tmpi++)
{
calculateZ(&res[tmpi],
a0+idx_fg, a1, a2, a3,
a4, a5, a6, a7,
a8, a9, a10, a11,
a12, a13, a14, a15,
tmpi, MOD);
}
printf("a=[%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d,], res=[%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,] Y=[%d %d %d %d %d %d %d %d %d %d]\n",
a0+idx_fg, a1, a2, a3,
a4, a5, a6, a7,
a8, a9, a10, a11,
a12, a13, a14, a15,
res[0], res[1], res[2], res[3],
res[4], res[5], res[6], res[7],
res[8], res[9], res[10], res[11],
res[12], res[13], res[14], res[15],
Y[0], Y[1], Y[2], Y[3], Y[4],
Y[5], Y[6], Y[7], Y[8], Y[9]);
}

}

}
// Now we have 8 distinct values, check each value
//if (Y[0]==1 && Y[1]==1 && Y[4]==1 && Y[6]==1 && Y[8]==1 && Y[9]==1 && Y[12]==1 && Y[13]==1)
//    if (Y[0] && Y[1] && Y[4] && Y[6] && Y[8] && Y[9] && Y[12] && Y[13])
//    {
//            // Below method can print inter-cross.
//            //for (int i=0;i<20;i++) printf("%d ", Y[i]);
//            //printf("a=%d, b=%d, c=%d, d=%d, e=%d, f=%d\n", a,b,c,d,e,f);
//            printf("a=[%2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d %2d], Y=[%d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d ]\n",
//            a0, a1, a2, a3,
//            a4, a5, a6, a7,
//            a8, a9, a10, a11,
//            a12, a13, a14, a15,
//
//    Y[0], Y[1], Y[2], Y[3], Y[4],
//    Y[5], Y[6], Y[7], Y[8], Y[9],
//    Y[10], Y[11], Y[12], Y[13], Y[14],
//    Y[15], Y[16], Y[17], Y[18], Y[19]);
//Y[18]++;
//    }

}