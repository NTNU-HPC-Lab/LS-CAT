#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__global__ void AplusB(int *ret, int a, int N)
{
size_t gindex = threadIdx.x + blockIdx.x * blockDim.x;
if ( gindex < N ) // Only change the needed.
ret[gindex] = a + gindex;
}