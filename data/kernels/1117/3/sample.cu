#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__global__ void sample()
{
int a = blockIdx.x;
int b = blockIdx.y;
int c = threadIdx.x;
double x = 1;

double result = pow(0.0,x)+a+b*x+c*pow(x,2.0);

if(result == 10)
printf("a=%d, b=%d, c=%d\n", a,b,c);
}