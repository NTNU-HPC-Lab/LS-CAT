#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__global__ void vDisp(const float *A, const float *B, int ds)
{
int idx = blockIdx.x * block_size + threadIdx.x; // create typical 1D thread index from built-in variables
printf("idx = %d, ds = %d\n", idx, ds);
if (idx < ds)
printf("Device: [%d], \t%f\t%f \n", idx, A[idx], B[idx]);         // do the vector (element) add here
}