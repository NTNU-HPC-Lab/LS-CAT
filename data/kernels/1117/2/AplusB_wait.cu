#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__global__ void AplusB_wait(int *ret, int a, int N, clock_t sleepInterval)
{
clock_t start = clock64();
while ( clock64() < start + sleepInterval ) { }
size_t gindex = threadIdx.x + blockIdx.x * blockDim.x;
if ( gindex < N ) // Only change the needed.
ret[gindex] = a + gindex;
}