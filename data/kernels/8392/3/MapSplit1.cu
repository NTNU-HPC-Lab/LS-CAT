#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

/*
//åñëè äëÿ âñåõ êàðò õâàòèò âîçìîæíîñòåé âèäåîêàðòû (ÐÀÁÎÒÀÅÒ)
*/






__global__ void MapSplit1(const int* one, int* result, unsigned int mx, unsigned int width)
{
const unsigned int ppp = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int rix = ppp % width;
const unsigned int riy = (ppp / mx) + ((ppp % mx) / width);
const unsigned int xxx = riy * width + rix;
const unsigned int ddx = riy * mx + rix;
result[xxx] = one[ddx];
}