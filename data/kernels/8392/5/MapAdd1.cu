#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

/*
//åñëè äëÿ âñåõ êàðò õâàòèò âîçìîæíîñòåé âèäåîêàðòû (ÐÀÁÎÒÀÅÒ)
*/






__global__ void MapAdd1(int* one, const int* result, unsigned int mx, unsigned int width)
{
const unsigned int ppp = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int rix = ppp % width;
const unsigned int riy = (ppp / mx) + ((ppp % mx) / width);
const unsigned int xxx = riy * width + rix;
const unsigned int ddx = riy * mx + rix;
one[ddx] = result[xxx];
}