#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
}