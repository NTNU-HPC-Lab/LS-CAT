#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;

x[index] = x[index] - (rate * sqrt(1.-pow(B2, t)) / (1.-pow(B1, t)) * m[index] / (sqrt(v[index]) + eps));
//if(index == 0) printf("%f %f %f %f\n", m[index], v[index], (rate * sqrt(1.-pow(B2, t)) / (1.-pow(B1, t)) * m[index] / (sqrt(v[index]) + eps)));
}