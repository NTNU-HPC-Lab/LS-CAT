#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= n) return;

int k = id % c;
id /= c;
int b = id;

int i;
int out_index = (k + c*b);
for(i = 0; i < w*h; ++i){
int in_index = i + h*w*(k + b*c);
in_delta[in_index] += out_delta[out_index] / (w*h);
}
}