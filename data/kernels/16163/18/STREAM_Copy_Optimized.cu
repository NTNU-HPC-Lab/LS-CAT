#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Copy_Optimized(float *a, float *b, size_t len)
{
/*
* Ensure size of thread index space is as large as or greater than
* vector index space else return.
*/
if (blockDim.x * gridDim.x < len) return;
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < len) b[idx] = a[idx];
}