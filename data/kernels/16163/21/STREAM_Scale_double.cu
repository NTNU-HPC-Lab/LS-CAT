#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale_double(double *a, double *b, double scale,  size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
b[idx] = scale* a[idx];
idx   += blockDim.x * gridDim.x;
}
}