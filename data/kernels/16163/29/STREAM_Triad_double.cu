#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad_double(double *a, double *b, double *c, double scalar, size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
c[idx] = a[idx]+scalar*b[idx];
idx   += blockDim.x * gridDim.x;
}
}