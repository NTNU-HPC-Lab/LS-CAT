#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale(float *a, float *b, float scale,  size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
b[idx] = scale* a[idx];
idx   += blockDim.x * gridDim.x;
}
}