#include "hip/hip_runtime.h"
#include "includes.h"



#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void maxpooling(float *output, const float *input, const int width, const int channels)
{
int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
int new_width = width / 2;
int i = thread_id / new_width * 2;
int j = thread_id % new_width * 2;
int index = i * width + j;

for (int c = 0; c < channels; c++) {
float max = 0;
if (max < input[index * channels + c])
max = input[index * channels + c];
if (max < input[(index + 1) * channels + c])
max = input[(index + 1) * channels + c];
if (max < input[(index + width) * channels + c])
max = input[(index + width) * channels + c];
if (max < input[(index + width + 1) * channels + c])
max = input[(index + width + 1) * channels + c];
output[thread_id * channels + c] = max;
}
}