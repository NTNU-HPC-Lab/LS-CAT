#include "hip/hip_runtime.h"
#include "includes.h"



#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void transform_image(float *input, const float *raw_input, const int width, const int channels)
{
int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
int start_i = thread_id / width - 1;
int start_j = thread_id % width - 1;
int per_channel_width = width * width;
int hidden_width = 3 * 3 * channels + 1;
int global_offset = thread_id * hidden_width;

for (int c = 0; c < channels; c++) {
int offset = 0;
for (int i = start_i; i < start_i + 3; i++) {
if (i < 0 || i == width)
continue;
for (int j = start_j; j < start_j + 3; j++) {
if (j < 0 || j == width)
continue;
input[global_offset + c * 9 + offset] = raw_input[c * per_channel_width + i * width + j];
offset++;
}
}
}
input[(thread_id + 1) * hidden_width - 1] = 1;
}