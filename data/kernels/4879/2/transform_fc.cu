#include "hip/hip_runtime.h"
#include "includes.h"



#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void transform_fc(float *input, const float *raw_input, const int width, const int channels)
{
int thread_id = threadIdx.x;
int size = width * width;

for (int s = 0; s < size; s++)
input[thread_id * size + s] = raw_input[s * channels + thread_id];
if (thread_id == 0)
input[width * width * channels] = 1;
}