#include "hip/hip_runtime.h"
#include "includes.h"



#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void transform(float *input, const float *raw_input, const int width, const int channels)
{
int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
int start_i = thread_id / width - 1;
int start_j = thread_id % width - 1;
int hidden_width = 3 * 3 * channels + 1;
int global_offset = thread_id * hidden_width;

float relu;
for (int c = 0; c < channels; c++) {
int offset = 0;
for (int i = start_i; i < start_i + 3; i++) {
if (i < 0 || i == width)
continue;
for (int j = start_j; j < start_j + 3; j++) {
if (j < 0 || j == width)
continue;
relu = raw_input[(i * width + j) * channels + c];
input[global_offset + c * 9 + offset] = relu < 0 ? 0 : relu;
offset++;
}
}
}
input[(thread_id + 1) * hidden_width - 1] = 1;
}