#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16


__global__ void gameOfLifeKernel(unsigned char* d_src, unsigned char* d_dst, const size_t width, const size_t height) {
extern __shared__ unsigned char board_sh[];

size_t glob_x = blockDim.x * blockIdx.x + threadIdx.x;
size_t glob_y = blockDim.y * blockIdx.y + threadIdx.y;
size_t glob_idx = glob_y * width + glob_x;

size_t index = blockDim.x * threadIdx.y + threadIdx.x;

int share_width = blockDim.x + 2;
int share_height = blockDim.y + 2;
int share_size = share_width * share_height;

bool isActive = (glob_x < width && glob_y < height);

// Set shared memory
for (int share_idx = index; share_idx < share_size; share_idx += (blockDim.x * blockDim.y)) {

int x_img = (blockDim.x * blockIdx.x - 1) + (share_idx % share_width);
int y_img = (blockDim.y * blockIdx.y - 1) + (share_idx / share_width);

if (x_img < 0) {
x_img = width - 1;
} else if (x_img > width - 1) {
x_img = 0;
}

if (y_img < 0) {
y_img = height - 1;
} else if (y_img > height - 1) {
y_img = 0;
}

board_sh[share_idx] = d_src[width * y_img + x_img];
}

__syncthreads();

if (isActive) {

unsigned char me = board_sh[share_width * (threadIdx.y + 1) + threadIdx.x + 1];

// Count neighbors
int count = board_sh[share_width * (threadIdx.y) + threadIdx.x];
count += board_sh[share_width * (threadIdx.y) + threadIdx.x + 1];
count += board_sh[share_width * (threadIdx.y) + threadIdx.x + 2];
count += board_sh[share_width * (threadIdx.y + 1) + threadIdx.x];
count += board_sh[share_width * (threadIdx.y + 1) + threadIdx.x + 2];
count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x];
count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x + 1];
count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x + 2];

// Game of life rules
if (me == 1) {
if (count < 2) {
d_dst[glob_idx] = 0;
}
else if (count < 4) {
d_dst[glob_idx] = 1;
}
else {
d_dst[glob_idx] = 0;
}
} else {
if (count == 3) {
d_dst[glob_idx] = 1;
}
}
}
}