#include "hip/hip_runtime.h"
#include "includes.h"

#pragma diag_suppress integer_sign_change

static unsigned short* d_in;
static unsigned char* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 12
#define THREAD_AUX_X_LEN 4
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 12
#define THREAD_AUX_Y_LEN 4
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x


__global__ void Sobel(const unsigned short* in,unsigned char* out, const unsigned int width, const unsigned int height)
{
extern __shared__ float s[];

const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
const unsigned int inPos = (xPos + yPos * width);
const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);
unsigned int outIt = inPos * 4;

if (xPos < width && yPos < height)
s[sIdx] = in[inPos] / float(USHRT_MAX);
else
s[sIdx] = 0.0f;

__syncthreads();

if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_Y_LEN)
{
const float sobelX = (
-1 * s[OFFSET(-2,-2)] -2 * s[OFFSET(-1,-2)] +0 * s[OFFSET(0,-2)] +2 * s[OFFSET(1,-2)] +1 * s[OFFSET(2,-2)]
-1 * s[OFFSET(-2,-1)] -2 * s[OFFSET(-1,-1)] +0 * s[OFFSET(0,-1)] +2 * s[OFFSET(1,-1)] +1 * s[OFFSET(2,-1)]
-2 * s[OFFSET(-2, 0)] -4 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +4 * s[OFFSET(1, 0)] +2 * s[OFFSET(2, 0)]
-1 * s[OFFSET(-2, 1)] -2 * s[OFFSET(-1, 1)] +0 * s[OFFSET(0, 1)] +2 * s[OFFSET(1, 1)] +1 * s[OFFSET(2, 1)]
-1 * s[OFFSET(-2, 2)] -2 * s[OFFSET(-1, 2)] +0 * s[OFFSET(0, 2)] +2 * s[OFFSET(1, 2)] +1 * s[OFFSET(2, 2)]
)*(512/8)/18;

const float sobelY =	(
+1 * s[OFFSET(-2,-2)] +1 * s[OFFSET(-1,-2)] +2 * s[OFFSET(0,-2)] +1 * s[OFFSET(1,-2)] +1 * s[OFFSET(2,-2)]
+2 * s[OFFSET(-2,-1)] +2 * s[OFFSET(-1,-1)] +4 * s[OFFSET(0,-1)] +2 * s[OFFSET(1,-1)] +2 * s[OFFSET(2,-1)]
+0 * s[OFFSET(-2, 0)] +0 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +0 * s[OFFSET(1, 0)] +0 * s[OFFSET(2, 0)]
-2 * s[OFFSET(-2, 1)] -2 * s[OFFSET(-1, 1)] -4 * s[OFFSET(0, 1)] -2 * s[OFFSET(1, 1)] -2 * s[OFFSET(2, 1)]
-1 * s[OFFSET(-2, 2)] -1 * s[OFFSET(-1, 2)] -2 * s[OFFSET(0, 2)] -1 * s[OFFSET(1, 2)] -1 * s[OFFSET(2, 2)]
)*(512/8)/18;

const float gradientLen = sqrt(sobelX*sobelX + sobelY*sobelY + 1.0f);

const unsigned char xLen = -(sobelX * 128)/gradientLen + 128;
const unsigned char yLen = -(sobelY * 128)/gradientLen + 128;
const unsigned char zLen = (UCHAR_MAX)/gradientLen;

out[outIt++] = xLen;
out[outIt++] = yLen;
out[outIt++] = zLen;
out[outIt] = 255;
}
}