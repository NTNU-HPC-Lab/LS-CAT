#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aypb_f32 (float a, float* y, float b, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] = a * y[idx] + b;
}
}