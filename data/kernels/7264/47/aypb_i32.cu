#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aypb_i32 (int a, int* y, int b, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] = a * y[idx] + b;
}
}