#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution(float* input, int inputRows, int inputCols, int inputLd, float* kernel, int kernelRows, int kernelCols, int kernelLd, int rowStep, int colStep, float* output, int outputLd) {

int row = (blockIdx.y * blockDim.y + threadIdx.y) * rowStep;
int col = (blockIdx.x * blockDim.x + threadIdx.x) * colStep;

if (row <= inputRows - kernelRows && col <= inputCols - kernelCols) {
int i, j;
output[row+col*outputLd] = 0;
for (i=0; i<kernelRows; i++) {
for (j=0; j<kernelCols; j++) {
output[row+col*outputLd] += kernel[i+j*kernelLd] * input[(row+i)+(col+j)*inputLd];
}
}
}

}