#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xvpy_f32 (float* x, float* v, float* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] += x[idx] * v[idx];
}
}