#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void customErrorCalc_f32 (float* vector, float* ideal_vector, float threshold, float scaleFoff, float scaleFon, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
float vectorValue = vector[idx];
if (ideal_vector[idx] > threshold) {
output[idx] = 1.0 - vectorValue;
if (vectorValue < threshold) {
output[idx] *= scaleFoff;
}
} else {
output[idx] = vectorValue * vectorValue;
if (vectorValue > threshold) {
output[idx] *= scaleFon;
}
}
}
}