#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpb_y_i32 (int a, int* x, int b, int* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] *= a * x[idx] + b;
}
}