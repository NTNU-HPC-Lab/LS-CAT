#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpb_y_f32 (float a, float* x, float b, float* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] *= a * x[idx] + b;
}
}