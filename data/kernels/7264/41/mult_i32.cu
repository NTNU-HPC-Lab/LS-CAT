#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_i32 (int* left_op, int* right_op, int* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = left_op[idx] * right_op[idx];
}
}