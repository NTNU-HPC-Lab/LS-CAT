#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_f32 (float* vector, float value, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
vector[idx] = value;
}
}