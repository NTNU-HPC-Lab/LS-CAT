#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_i32 (int* vector, int* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = vector[idx] * vector[idx];
}
}