#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x_avpb_py_i32 (int* x, int a, int* v, int b, int* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] += x[idx] * (a * v[idx] + b);
}
}