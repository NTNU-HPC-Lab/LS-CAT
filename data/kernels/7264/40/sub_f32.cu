#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_f32 (float* left_op, float* right_op, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = left_op[idx] - right_op[idx];
}
}