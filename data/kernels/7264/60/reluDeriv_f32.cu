#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluDeriv_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = vector[idx] > 0.0 ? 1.0 : 0.0;
}
}