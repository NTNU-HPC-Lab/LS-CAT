#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoidDeriv_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
float tmp = 1.0 + (vector[idx] < 0.0 ? -vector[idx] : vector[idx]);   output[idx] = - 0.5 / (tmp*tmp);
}
}