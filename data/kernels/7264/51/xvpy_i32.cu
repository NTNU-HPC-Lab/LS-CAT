#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xvpy_i32 (int* x, int* v, int* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] += x[idx] * v[idx];
}
}