#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_f32 (float* vector, float threshold, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = vector[idx] > threshold ? 1 : 0;
}
}