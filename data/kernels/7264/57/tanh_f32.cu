#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanh_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
float tmp = vector[idx];   output[idx] = tmp / (1.0 + (tmp < 0.0 ? -tmp : tmp));
}
}