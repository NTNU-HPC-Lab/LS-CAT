#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanhDeriv_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
float tmp = vector[idx] < 0.0 ? -vector[idx] : vector[idx];   output[idx] =  1.0 / ((1.0+tmp)*(1.0+tmp));
}
}