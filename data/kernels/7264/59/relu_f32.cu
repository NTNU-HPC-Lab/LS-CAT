#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
output[idx] = vector[idx] > 0.0 ? vector[idx] : 0.0;
}
}