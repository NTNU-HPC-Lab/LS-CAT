#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x_avpb_py_f32 (float* x, float a, float* v, float b, float* y, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
y[idx] += x[idx] * (a * v[idx] + b);
}
}