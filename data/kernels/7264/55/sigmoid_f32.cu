#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid_f32 (float* vector, float* output, int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) {
float tmp = vector[idx];   output[idx] = 0.5 - 0.5 * tmp / (1.0 + (tmp < 0.0 ? -tmp : tmp));
}
}