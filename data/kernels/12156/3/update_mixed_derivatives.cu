#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_mixed_derivatives(double *temppsix, double *temppsiy, double *temppsixy, unsigned int nx, unsigned int ny, double dx, double dy, unsigned int TileSize)
{
unsigned int bx = blockIdx.x;
unsigned int by = blockIdx.y;

unsigned int tx = threadIdx.x;
unsigned int ty = threadIdx.y;

unsigned int index_x = bx * TileSize + tx;
unsigned int index_y = by * TileSize + ty;

unsigned int indexToWrite = index_y * nx + index_x;

if ((index_y == 0 || index_y == ny - 1) && (index_x != 0 && index_x != nx - 1))
temppsixy[indexToWrite] = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1])/(2 * dx);
else
if ((index_y != 0 && index_y != ny - 1) && (index_x == 0 || index_x == nx - 1))
temppsixy[indexToWrite] = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx])/(2 * dy);
else
if((index_y == 0 || index_y == ny - 1) && (index_x == 0 || index_x == nx - 1)){
if(index_y == 0 && index_x == 0){
double d1 = (temppsiy[1] - temppsiy[0])/dx;
double d2 = (temppsix[nx] - temppsix[0])/dy;
double d3 = (temppsix[nx+1] - temppsix[1])/dy;
double d4 = (temppsiy[nx+1] - temppsiy[nx])/dx;
temppsixy[indexToWrite] = 0.75 * (d1 + d2) - 0.25 * (d3 + d4);
}
else if(index_y == 0 && index_x == nx-1){
double d1 = (temppsiy[nx-1] - temppsiy[nx-2])/dx;
double d2 = (temppsix[nx+nx-2] - temppsix[nx-2])/dy;
double d3 = (temppsix[nx+nx-1] - temppsix[nx-1])/dy;
double d4 = (temppsiy[nx+nx-1] - temppsiy[nx+nx-2])/dx;
temppsixy[indexToWrite] = 0.75 * (d1 + d3) - 0.25 * (d2 + d4);

}
else if(index_y == ny-1 && index_x == 0){
double d1 = (temppsiy[nx *(ny-2) + 1] - temppsiy[nx *(ny-2)])/dx;
double d2 = (temppsix[nx *(ny-1)] - temppsix[nx *(ny-2)])/dy;
double d3 = (temppsix[nx *(ny-1)] - temppsix[nx *(ny-2) + 1])/dy;
double d4 = (temppsiy[nx *(ny-1) + 1] - temppsiy[nx *(ny-1)])/dx;
temppsixy[indexToWrite] = 0.75 * (d2 + d4) - 0.25 * (d3 + d1);

}
else if(index_y == ny-1 && index_x == nx-1){
double d1 = (temppsiy[nx *(ny-2) + nx - 1] - temppsiy[nx *(ny-2) + nx - 2])/dx;
double d2 = (temppsix[nx *(ny-1) + nx - 2] - temppsix[nx *(ny-2) + nx - 2])/dy;
double d3 = (temppsix[nx *(ny-1) + nx - 1] - temppsix[nx *(ny-2) + nx - 1])/dy;
double d4 = (temppsiy[nx *(ny-1) + nx - 1] - temppsiy[nx *(ny-1) + nx - 2])/dx;
temppsixy[indexToWrite] = 0.75 * (d3 + d4) - 0.25 * (d1 + d2);
}
}
else{
double dxy1 = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1])/(2 * dx);
double dxy2 = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx])/(2 * dy);
temppsixy[indexToWrite] = (dxy1 + dxy2)/2.0;
}

}