#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuImageBrighten(const float *dev_image, float *dev_out, int w, int h)
{
int tx = threadIdx.x;   int ty = threadIdx.y;
int bx = blockIdx.x;	int by = blockIdx.y;

int pos = tx + 32*bx + w* ty + 32*w*by;
dev_out[pos] = min(255.0f, dev_image[pos] + 50);
__syncthreads();
}