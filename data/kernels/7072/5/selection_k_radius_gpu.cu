#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_k_radius_gpu(int b, int m, int k, float radius, const int* idx, const float* val, int* idx_out, float* val_out){
int batch_index = blockIdx.x;
int stride = batch_index * m * k;
idx += stride;
val += stride;
idx_out += stride;
val_out += stride;
for(int i = threadIdx.x; i < m;i += blockDim.x) {

for(int j = 0;j < k;j ++) {
if(val[i * k + j] < radius) {
idx_out[i * k + j] = idx[i * k + j];
val_out[i * k + j] = val[i * k + j];
} else {
idx_out[i * k + j] = idx[i * k ];
val_out[i * k + j] = val[i * k ];
}
}
}
}