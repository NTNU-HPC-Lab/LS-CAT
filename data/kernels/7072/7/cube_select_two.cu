#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube_select_two(int b, int n,float radius, const float* xyz, int* idx_out) {
int batch_idx = blockIdx.x;
xyz += batch_idx * n * 3;
idx_out += batch_idx * n * 16;
float temp_dist[16];
float judge_dist = radius * radius;
for(int i = threadIdx.x; i < n;i += blockDim.x) {
float x = xyz[i * 3];
float y = xyz[i * 3 + 1];
float z = xyz[i * 3 + 2];
for(int j = 0;j < 16;j ++) {
temp_dist[j] = judge_dist;
idx_out[i * 16 + j] = i; // if not found, just return itself..
}
for(int j = 0;j < n;j ++) {
if(i == j) continue;
float tx = xyz[j * 3];
float ty = xyz[j * 3 + 1];
float tz = xyz[j * 3 + 2];
float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
if(dist > judge_dist) continue;
int _x = (tx > x);
int _y = (ty > y);
int _z = (tz > z);
int temp_idx = _x * 8 + _y * 4 + _z * 2;
bool flag = false;
for(int k = 0;k < 2;k ++) {
if (dist < temp_dist[temp_idx + k]) {
flag = true;
}
if (flag) {
for (int kk = 1; kk >= k + 1; kk --) {
idx_out[i * 16 + temp_idx + kk] = idx_out[i * 16 + temp_idx + kk - 1];
temp_dist[temp_idx + kk] = temp_dist[temp_idx + kk - 1];
}
idx_out[i * 16 + temp_idx + k] = j;
temp_dist[temp_idx + k] = dist;
break;
}
}

}
}
}