#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube_select(int b, int n,float radius, const float* xyz, int* idx_out) {
int batch_idx = blockIdx.x;
xyz += batch_idx * n * 3;
idx_out += batch_idx * n * 8;
float temp_dist[8];
float judge_dist = radius * radius;
for(int i = threadIdx.x; i < n;i += blockDim.x) {
float x = xyz[i * 3];
float y = xyz[i * 3 + 1];
float z = xyz[i * 3 + 2];
for(int j = 0;j < 8;j ++) {
temp_dist[j] = 1e8;
idx_out[i * 8 + j] = i; // if not found, just return itself..
}
for(int j = 0;j < n;j ++) {
if(i == j) continue;
float tx = xyz[j * 3];
float ty = xyz[j * 3 + 1];
float tz = xyz[j * 3 + 2];
float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
if(dist > judge_dist) continue;
int _x = (tx > x);
int _y = (ty > y);
int _z = (tz > z);
int temp_idx = _x * 4 + _y * 2 + _z;
if(dist < temp_dist[temp_idx]) {
idx_out[i * 8 + temp_idx] = j;
temp_dist[temp_idx] = dist;
}
}
}
}