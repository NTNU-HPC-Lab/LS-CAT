#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayReduce(int *m, int *ms){
int id = threadIdx.x + blockIdx.x * blockDim.x;
if (m[id] > -1)
m[id] = m[id] - ms[blockIdx.x];
}