#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel(int *s, const int *re){

__shared__ int temp[1];

int i = threadIdx.x;
if (re[i] > -1 && re[i] < temp[0])
temp[0] = re[i];

__syncthreads();

*s = temp[0];
}