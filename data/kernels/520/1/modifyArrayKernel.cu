#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modifyArrayKernel(int *val, int *arr){
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < 6 && arr[i] > -1)
arr[i] = arr[i] - *val;
}