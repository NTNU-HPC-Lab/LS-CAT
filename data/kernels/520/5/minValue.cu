#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minValue(int *source, int *val){
__shared__ int temp[1];

int currentValue = source[threadIdx.x];
if (currentValue > -1 && currentValue < *val){
temp[0] = currentValue;
}

__syncthreads();

*val = temp[0];
}