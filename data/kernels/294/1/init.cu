#include "hip/hip_runtime.h"
#include "includes.h"

const int CHUNKS = 64;
const int GENERATIONS = 10;


const int CHECK_VALUES_EVERY = 50000;
const int SHOW_ALL_VALUES = 0;
const int SKIP_CUDA_DEVICE = false;


const int ISLANDS_PER_ROW = 4;
const int GENOME_LENGTH=4;
const int BLOCKS_PER_ROW = 4;
const int ISLAND_POPULATION=100;
const int SELECTION_COUNT=80;
const float MUTATION_CHANCE= 0.8;
const int ITEMS_MAX_WEIGHT = 5;
const int ITEMS_MAX_VALUE = 20;
const int ITEMS_MAX = 20;

__global__ void init(unsigned int seed, hiprandState_t* states) {

/* we have to initialize the state */
hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
blockDim.y * blockIdx.y , /* the sequence number should be different for each core (unless you want all
cores to get the same sequence of numbers for some reason - use thread id! */
0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&states[blockDim.y * blockIdx.y ]);
}