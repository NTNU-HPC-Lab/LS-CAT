#include "hip/hip_runtime.h"
#include "includes.h"

const int CHUNKS = 64;
const int GENERATIONS = 10;


const int CHECK_VALUES_EVERY = 50000;
const int SHOW_ALL_VALUES = 0;
const int SKIP_CUDA_DEVICE = false;


const int ISLANDS_PER_ROW = 4;
const int GENOME_LENGTH=4;
const int BLOCKS_PER_ROW = 4;
const int ISLAND_POPULATION=100;
const int SELECTION_COUNT=80;
const float MUTATION_CHANCE= 0.8;
const int ITEMS_MAX_WEIGHT = 5;
const int ITEMS_MAX_VALUE = 20;
const int ITEMS_MAX = 20;

__global__ void randomizePopulation(hiprandState_t* states, unsigned char* population ) {
int island_y = blockDim.y * blockIdx.y + threadIdx.y;
int island_x = blockDim.x * blockIdx.x + threadIdx.x;

__shared__ hiprandState_t randomState;
randomState = states[blockDim.y * blockIdx.y ];

unsigned char * populationRow = &population[island_y * GENOME_LENGTH * ISLAND_POPULATION * ISLANDS_PER_ROW + island_x * GENOME_LENGTH * ISLAND_POPULATION ];

for(int i = 0; i < GENOME_LENGTH * ISLAND_POPULATION; i++) {
populationRow[i] = hiprand(&randomState) % ITEMS_MAX;
};
}