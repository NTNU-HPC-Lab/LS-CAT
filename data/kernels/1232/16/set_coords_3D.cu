#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_coords_3D(float* coords, size_t z, size_t y, size_t x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t id_x = index % x;
size_t id_y = (index / x) % y;
size_t id_z = index / (x * y);
if(index < x * y * z){
coords[index] = id_z - (float)z/2.0;
coords[index + x * y * z] = id_y - (float)y/2.0;
coords[index + 2 * x * y * z] = id_x -(float)x/2.0;
}
__syncthreads();
}