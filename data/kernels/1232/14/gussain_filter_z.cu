#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ int mirror(int index, int len){
int s2 = 2 * len - 2;
if(index < 0){
index = s2 * (-index / s2) + index;
return index <= 1 - len ? index + s2 : -index;
}
if(index >= len){
index -= s2 * (index / s2);
if(index >= len)
index = s2 - index;
return index;
}
if(index < 0 || index >= len) index = mirror(index, len);
return index;
}
__global__ void gussain_filter_z(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y * dim_z;
size_t total_xy = dim_x * dim_y;
size_t id_x = index % dim_x;
size_t id_y = (index / dim_x) % dim_y;
size_t id_z = (index / total_xy) % dim_z;
size_t id_block = index / total;
int id;
float new_pixel = 0;
int dim = 2;
if(dim_z > 1){
dim = 3;
}
if(index < total * dim){
if(mode == 0){
for(int i = -lw; i < lw + 1; i++){
id = id_z + i;
if(id < 0 || id > dim_z - 1)
new_pixel += cval * kernel[i+lw];
else new_pixel += kernel[i+lw] *
random[id_block * total + id * total_xy + id_y * dim_x + id_x];
}
__syncthreads();
random[index] = new_pixel;
__syncthreads();
}
else{
for(int i = -lw; i < lw + 1; i++){
id = id_z + i;
id = mirror(id, id_z);
new_pixel += kernel[i+lw] *
random[id_block * total + id * total_xy + id_y * dim_x + id_x];
}
__syncthreads();
random[index] = new_pixel;
__syncthreads();
}
}
}