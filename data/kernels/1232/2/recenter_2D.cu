#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void recenter_2D(float* coords, size_t dim_y, size_t dim_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < dim_x * dim_y){
coords[index] += (float)dim_y/2.0;
coords[index + dim_x*dim_y] += (float)dim_x/2.0;
}
__syncthreads();
}