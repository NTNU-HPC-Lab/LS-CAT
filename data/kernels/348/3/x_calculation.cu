#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);





__global__ void x_calculation(float * x ,float * r,float * r_squared ,int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x ;
if (index < size)
{
float alpha = r_squared[0] ;
x[index] = x[index] + alpha * r[index] ;
}
}