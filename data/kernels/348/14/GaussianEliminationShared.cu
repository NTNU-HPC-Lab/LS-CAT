#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianEliminationShared(const int clusterSize,float *x, const float *diagonal_values , const float *non_diagonal_values ,float *y )
{
const int index = blockIdx.x ;

__shared__ float shared_m[9][9]; // size of cluster
for (int i = 0; i < clusterSize;++i)
{
for (int j = 0; j < clusterSize;++j)
{
shared_m[i][j]=0;
}
}
for(int i = 0; i < clusterSize; ++i)
{
shared_m[i][i] = diagonal_values[clusterSize * index + i];
}
for(int i = 0; i < clusterSize-1;++i)
{
shared_m[i][i+1] = non_diagonal_values[clusterSize * index * 2 + 2*i+1];
shared_m[i+1][i] = non_diagonal_values[clusterSize * index * 2 + 2*i+2];
}

// triangle form
for (int i = 0 ; i < clusterSize; ++i)
{
//for every row...
for (int j = i+1; j < clusterSize; ++j)
{
//calculate ratio for every row below it using the triangular
double ratio = shared_m[j][i] / shared_m[i][i];
for(int k = 0; k < clusterSize; ++k)
{
//Eliminate every column based on that ratio
shared_m[j][k] = shared_m[j][k] - (shared_m[i][k] * ratio);
}
//elimination on the coefficient vector
y[clusterSize * index +j] = y[clusterSize * index +j] - (y[clusterSize * index +i] * ratio);
}
}
__syncthreads();

//Back substitution
for (int i = clusterSize-1; i > -1; --i)
{
double current = 0;
for (unsigned int j = i; j < clusterSize; ++j)
{
current = current + (shared_m[i][j] * x[clusterSize * index +j]);
}
x[clusterSize * index +i] = (y[clusterSize * index +i] - current) / shared_m[i][i];
}

}