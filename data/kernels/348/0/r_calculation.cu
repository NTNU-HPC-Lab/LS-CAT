#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);





__global__ void r_calculation(float* a , int * indeces , float* b , float* x,float * r  ,int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x ;

if (index < size)
{
float sum = 0 ;

for (int i = 0 ; i<3 ; i++)
{
sum += a[3*index  + i] * x[indeces[3*index + i]] ;
}

r[index] = b[index] - sum ;
}

}