#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divide(float *x, float* y ,float* out ,const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size)
{
out[index] = x[index]/y[index] ;
}
}