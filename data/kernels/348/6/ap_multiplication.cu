#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ap_multiplication(float * values ,int * indeces,float* r ,float * p_sum ,int size)
{
int index = blockDim.x * blockIdx.x + threadIdx.x ;

p_sum[index] = 0;
__syncthreads() ;
if (index < size)
{
for (int i = 0 ; i<3 ; i++)
{
p_sum[index] += values[3*index  + i] * r[indeces[3*index + i]] ;
}
__syncthreads() ;
}
}