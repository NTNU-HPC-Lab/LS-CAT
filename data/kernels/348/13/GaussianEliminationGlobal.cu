#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianEliminationGlobal(const int clusterSize,float *x, const float *diagonal_values , const float *non_diagonal_values ,float *y , const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x ;
const int gi = index * clusterSize;

float matrix[180][180];	//size of matrix
for (int i = gi; i < gi + clusterSize;++i)
{
for (int j = gi; j < gi + clusterSize;++j)
{
matrix[i][j]=0;
}
matrix[i][i] = diagonal_values[i];
}
for(int i = gi; i < gi + clusterSize - 1 ;++i)
{
matrix[i][i+1] = non_diagonal_values[2*i+1];
matrix[i+1][i] = non_diagonal_values[2*i+2];
}

// triangle form
for (int i = gi ; i < gi + clusterSize; ++i)
{
//for every row...
for (int j = i+1; j < gi + clusterSize; ++j)
{
//calculate ratio for every row below it using the triangular
double ratio = matrix[j][i] / matrix[i][i];
for(int k = gi; k < gi + clusterSize; ++k)
{
//Eliminate every column based on that ratio
matrix[j][k] = matrix[j][k] - (matrix[i][k] * ratio);
}
//elimination on the coefficient vector
y[j] = y[j] - (y[i] * ratio);
}
}
__syncthreads();
//Back substitution
for (int i = gi + clusterSize-1; i > gi-1; --i)
{
double current = 0;
for (unsigned int j = i; j < gi + clusterSize; ++j)
{
current = current + (matrix[i][j] * x[j]);
}
x[i] = (y[i] - current) / matrix[i][i];
}
}