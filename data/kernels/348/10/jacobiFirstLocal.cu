#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobiFirstLocal(float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
const int index = blockIdx.x * blockDim.x + threadIdx.x;
float error = 1 ;
float current_value = 1 ;

if (index < size)
{
float local_diagonal_value ;
float local_non_diagonal_values[2];
int local_indeces[2];
float local_y;

local_diagonal_value = diagonal_values[index];
local_non_diagonal_values[0] = non_diagonal_values[2*index];
local_non_diagonal_values[1] = non_diagonal_values[2*index+1];
local_indeces[0] = indeces[2*index];
local_indeces[1] = indeces[2*index+1];
local_y = y[index];

float sum = 0 ;

while(fabsf(error) > 0.00001)
{
for (int i = 0 ; i<2 ; i++)
{
sum += local_non_diagonal_values[i]  * x[local_indeces[i]] ;
}

current_value = (local_y - sum )/local_diagonal_value;
error = current_value - x[index] ;
x[index] = current_value ;
sum = 0 ;
__syncthreads();
}
}
}