#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yuan(const char *text, int *pos, int text_size) {
int textP = blockIdx.x * blockDim.x + threadIdx.x;
if (textP >= text_size) return;
const char *start = text + textP;
while (start >= text && *start > ' ') {
start--;
}
pos[textP] = text + textP - start;

}