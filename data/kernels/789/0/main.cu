#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "loss_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int factors = 1;
int user_count = 1;
int item_count = 1;
const float *P = NULL;
hipMalloc(&P, XSIZE*YSIZE);
const float *Q = NULL;
hipMalloc(&Q, XSIZE*YSIZE);
const int *indptr = NULL;
hipMalloc(&indptr, XSIZE*YSIZE);
const int *indices = NULL;
hipMalloc(&indices, XSIZE*YSIZE);
const float *data = NULL;
hipMalloc(&data, XSIZE*YSIZE);
float *error = NULL;
hipMalloc(&error, XSIZE*YSIZE);
float *user_bias = NULL;
hipMalloc(&user_bias, XSIZE*YSIZE);
float *item_bias = NULL;
hipMalloc(&item_bias, XSIZE*YSIZE);
float global_bias = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
loss_kernel<<<gridBlock,threadBlock>>>(factors,user_count,item_count,P,Q,indptr,indices,data,error,user_bias,item_bias,global_bias);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
loss_kernel<<<gridBlock,threadBlock>>>(factors,user_count,item_count,P,Q,indptr,indices,data,error,user_bias,item_bias,global_bias);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
loss_kernel<<<gridBlock,threadBlock>>>(factors,user_count,item_count,P,Q,indptr,indices,data,error,user_bias,item_bias,global_bias);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}