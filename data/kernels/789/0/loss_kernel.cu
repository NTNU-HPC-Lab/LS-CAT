#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float get_prediction(int factors, const float *p, const float *q, float user_bias, float item_bias, float global_bias) {
float pred = global_bias + user_bias + item_bias;
for (int f = 0; f < factors; f++)
pred += q[f]*p[f];
return pred;
}
__global__ void loss_kernel(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, const int * indices, const float * data, float * error, float * user_bias, float * item_bias, float global_bias) {

// One thread per user
int u = blockDim.x * blockIdx.x + threadIdx.x;
if(u < user_count) {
// Get this user's factors and bias
const float * p = &P[u * factors];
const float ub = user_bias[u];

// Loop over all items of user
for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
int item_id = indices[i];
error[i] = data[i] - get_prediction(factors, p, &Q[item_id * factors], ub, item_bias[item_id], global_bias);
}
}
}