#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumaColMatrizKernel (int M, int N, float *Md, float *Nd){
// Pvalue es usado para el valor intermedio
__shared__ float Nds[DIMBLOCKX];
float Pvalue = 0;
int columna = blockIdx.y*(N/gridDim.x)+threadIdx.x;
int pasos = M/blockDim.x ;
int posIni = columna * M + threadIdx.x * pasos;
for (int k = 0; k < pasos; ++k) {
Pvalue = Pvalue + Md[posIni + k];
}
Nds[threadIdx.x] = Pvalue;
__syncthreads();
if (threadIdx.x == 0 ){
for (int i = 1; i < blockDim.x; ++i) {
Nds[0] = Nds[0]+Nds[i];
}

Nd[columna*gridDim.x+blockIdx.x] = Nds[0];

}
}