#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *r, int n, double gamma)
{
int i=threadIdx.x;

if(i<n)
r[i] = (int)(255.0*pow((double)a[i]/255.0,1.0/gamma));
}