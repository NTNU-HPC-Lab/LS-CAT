#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize_cells(CellT* dev_cells, CellT* dev_next_cells, int size_x, int size_y) {
for (int i = threadIdx.x + blockDim.x * blockIdx.x;
i < size_x*size_y; i += blockDim.x * gridDim.x) {
dev_cells[i] = 0;
dev_next_cells[i] = 0;
}
}