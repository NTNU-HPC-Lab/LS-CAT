#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bf_2flags(int *Na, int *src, int *F1, int *F2, int *exists, int *Sa, int *Ea, int threadsPerBlock )
{

int id = blockIdx.x * threadsPerBlock + threadIdx.x;

if (exists[id]==1)
{
Na[id] = 65000; //MAX INT Value
F1[id] = 0;
F2[id] = 0;

if (id == *src)
{	//Starting node conditions
Na[id] = 0;
F1[id] = 1;
}

for (int i = 0; i < 103689; ++i)
{
if (F1[Sa[id]] == 1)
{
if (Na[Ea[id]] > Na[Sa[id]] + 1)
{
//Relax
// atomicAdd(&Na[Ea[id]], Na[Sa[id]] + 1 - Na[Ea[id]]);
Na[Ea[id]] = Na[Sa[id]] + 1;
F2[Ea[id]] = 1;
}
}

//Swap flags
F1[id] = F2[id];
F2[id] = 0;
}
}
}