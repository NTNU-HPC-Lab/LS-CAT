#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianBlur(unsigned int *B,unsigned int *G,unsigned int *R, int numberOfPixels, unsigned int width, int *B_new, int *G_new, int *R_new)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= numberOfPixels){
//printf("%d\n",index);
return;
}

int mask[] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
int s = mask[0] + mask[1] + mask[2] + mask[3] + mask[4] + mask[5] + mask[6] + mask[7] + mask[8];

if (index < width){ // dolny rzad pikseli
if (index == 0){ //lewy dolny rog
s = mask[4] + mask[1] + mask[2] + mask[5];
B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5]) / s);
return;
}

if (index == width - 1){//prawy dolny rog
s = mask[4] + mask[0] + mask[1] + mask[3];
B_new[index] = (B[index] * mask[4] + B[index + width - 1] * mask[0] + B[index + width] * mask[1] + B[index - 1] * mask[3]);
G_new[index] = (G[index] * mask[4] + G[index + width - 1] * mask[0] + G[index + width] * mask[1] + G[index - 1] * mask[3]);
R_new[index] = (R[index] * mask[4] + R[index + width - 1] * mask[0] + R[index + width] * mask[1] + R[index - 1] * mask[3]);
return;
}
//reszta pikseli w dolnym rzedzie
s = mask[4] + mask[1] + mask[2] + mask[5] + mask[0] + mask[3];
B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index + width - 1] * mask[0] + B[index - 1] * mask[3]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5] + R[index + width - 1] * mask[0] + R[index - 1] * mask[3]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5] + G[index + width - 1] * mask[0] + G[index - 1] * mask[3]) / s);

return;
}
if (index >= numberOfPixels - width){ //gorny rzad pikseli

if (index == numberOfPixels - width){ //lewy gorny rog
s = mask[4] + mask[5] + mask[7] + mask[8];
B_new[index] = (int)((B[index] * mask[4] + B[index + 1] * mask[5] + B[index - width] * mask[7] + B[index - width + 1] * mask[8]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index + 1] * mask[5] + G[index - width] * mask[7] + G[index - width + 1] * mask[8]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index + 1] * mask[5] + R[index - width] * mask[7] + R[index - width + 1] * mask[8]) / s);
return;
}

if (index == numberOfPixels - 1){ //prawy gorny rog
s = mask[4] + mask[3] + mask[6] + mask[7];
B_new[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index - 1] * mask[3] + G[index - width - 1] * mask[6] + G[index - width] * mask[7]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index - 1] * mask[3] + R[index - width - 1] * mask[6] + R[index - width] * mask[7]) / s);
return;
}

s = mask[4] + mask[3] + mask[5] + mask[6] + mask[7] + mask[8];
B_new[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7] + B[index + 1] * mask[5] + B[index - width] * mask[8]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index - 1] * mask[3] + R[index - width - 1] * mask[6] + R[index - width] * mask[7] + R[index + 1] * mask[5] + R[index - width] * mask[8]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index - 1] * mask[3] + G[index - width - 1] * mask[6] + G[index - width] * mask[7] + G[index + 1] * mask[5] + G[index - width] * mask[8]) / s);
return;
}
if (index % width == 0){ //lewa sciana
s = mask[4] + mask[1] + mask[2] + mask[5] + mask[8] + mask[7];
B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index - width + 1] * mask[8] + B[index - width]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5] + G[index - width + 1] * mask[8] + G[index - width]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5] + R[index - width + 1] * mask[8] + R[index - width]) / s);
return;
}
if (index % width == width - 1){ //prawa sciana
s = mask[4] + mask[1] + mask[0] + mask[3] + mask[6] + mask[7];
B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width - 1] * mask[0] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7]) / s);
R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width - 1] * mask[0] + R[index - 1] * mask[3] + R[index - width - 1] * mask[6] + R[index - width] * mask[7]) / s);
G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width - 1] * mask[0] + G[index - 1] * mask[3] + G[index - width - 1] * mask[6] + G[index - width] * mask[7]) / s);
return;
}


int poz_1 = index - width - 1;
int poz_2 = index - width;
int poz_3 = index - width + 1;
int poz_4 = index - 1;
int poz_5 = index;
int poz_6 = index + 1;
int poz_7 = index + width - 1;
int poz_8 = index + width;
int poz_9 = index + width + 1;

B_new[index] = (int)(((B[poz_1] * mask[0]) + (B[poz_2] * mask[1]) + (B[poz_3] * mask[2]) + (B[poz_4] * mask[3]) + (B[poz_5] * mask[4]) + (B[poz_6] * mask[5]) + (B[poz_7] * mask[6]) + (B[poz_8] * mask[7]) + (B[poz_9] * mask[8])) / s);
G_new[index] = (int)(((G[poz_1] * mask[0]) + (G[poz_2] * mask[1]) + (G[poz_3] * mask[2]) + (G[poz_4] * mask[3]) + (G[poz_5] * mask[4]) + (G[poz_6] * mask[5]) + (G[poz_7] * mask[6]) + (G[poz_8] * mask[7]) + (G[poz_9] * mask[8])) / s);
R_new[index] = (int)(((R[poz_1] * mask[0]) + (R[poz_2] * mask[1]) + (R[poz_3] * mask[2]) + (R[poz_4] * mask[3]) + (R[poz_5] * mask[4]) + (R[poz_6] * mask[5]) + (R[poz_7] * mask[6]) + (R[poz_8] * mask[7]) + (R[poz_9] * mask[8])) / s);


}