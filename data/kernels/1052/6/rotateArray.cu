#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotateArray(int *c, int numThreads)
{
int nextIndex = (threadIdx.x + 1)%numThreads;
int val = c[nextIndex];

__syncthreads();

c[threadIdx.x] = val;
}