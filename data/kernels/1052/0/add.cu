#include "hip/hip_runtime.h"
#include "includes.h"
/*

Jaitirth Jacob - 13CO125      Vidit Bhargava - 13CO151

*/


#define ITERATIONS 4	//Repeat the experiment for greater accuracy


#define N 1000000		//Array Size
#define min_threads 16
#define max_threads 1024


__global__ void add(int *a, int *b, int *c, int tpb)
{
//Find the correct thread index in the grid
int i = blockIdx.x * tpb + threadIdx.x;
c[i] = a[i] + b[i];

}