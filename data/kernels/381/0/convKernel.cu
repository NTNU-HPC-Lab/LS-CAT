#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convKernel(const float* source, const float* kernel, float* target, const int len) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= len) return;

float value = 0.0f;

for (int i = 0; i < len; i++) {
value += source[i] *
kernel[(len + len / 2 + idx - i) % len]; // Positive modulo
}

target[idx] = value;
}