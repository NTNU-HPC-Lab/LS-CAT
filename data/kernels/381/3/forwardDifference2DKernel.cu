#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifference2DKernel(const int cols, const int rows, const float* data, float* dx, float* dy) {
for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1;
idy += blockDim.y * gridDim.y) {
for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
idx < rows - 1; idx += blockDim.x * gridDim.x) {
const auto index = idx + rows * idy;

dx[index] = data[index + 1] - data[index];
dy[index] = data[index + rows] - data[index];
}
}
}