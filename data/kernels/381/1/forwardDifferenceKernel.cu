#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifferenceKernel(const int len, const float* source, float* target) {
for (int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1;
idx += blockDim.x * gridDim.x) {
target[idx] = source[idx + 1] - source[idx];
}
}