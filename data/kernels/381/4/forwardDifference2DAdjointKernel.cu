#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows, const float* dx, const float* dy, float* target) {
for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1;
idy += blockDim.y * gridDim.y) {
for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
idx < rows - 1; idx += blockDim.x * gridDim.x) {
const auto index = idx + rows * idy;

target[index] =
-dx[index] + dx[index - 1] - dy[index] + dy[index - rows];
}
}
}