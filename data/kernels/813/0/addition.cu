#include "hip/hip_runtime.h"
#include "includes.h"
/* Addition of two numbers using a kernel method.
* Note: Documentation will explain each thing only once. */

/* Header files */


/* This a kernel function, it has the __global__ qualifier in the definition.
* addition: Perform the addition of two numbers and return their sum.
* +------------+-----------------------------------+
* | Parameters | Description                       |
* +------------+-----------------------------------+
* | int  a     | Takes an integer passed by value. |
* | int  b     | Takes an integer passed by value. |
* | int *c     | An integer pointer that refers to |
* |            | the GPU memory where we store the |
* |            | result of the addition.           |
* +------------+-----------------------------------+ */


/* main method runs on the host. */
__global__ void addition ( int a, int b, int *c )
{
*c = a + b;
}