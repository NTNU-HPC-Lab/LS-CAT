#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(const uint *__restrict__ a, const uint *__restrict__ b, const uint length, uint *c)
{
uint tid = (blockIdx.x * blockDim.x) + threadIdx.x;
const uint stride = blockDim.x * gridDim.x;
while (tid < length)
{
c[tid] = a[tid] + b[tid];
tid += stride;
}
}