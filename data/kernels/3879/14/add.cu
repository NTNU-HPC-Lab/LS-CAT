#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(const float3 *__restrict__ dFinalForce, const unsigned int noRainDrops, float3 *__restrict__ dRainDrops)
{
//TODO: Add the FinalForce to every Rain drops position.
uint xOffset = (blockIdx.x * blockDim.x) + threadIdx.x;
uint xSkip = gridDim.x * blockDim.x;

while (xOffset < noRainDrops)
{
dRainDrops[xOffset].x += dFinalForce->x;
dRainDrops[xOffset].y += dFinalForce->y;
dRainDrops[xOffset].z += dFinalForce->z;

xOffset += xSkip;
}
}