#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_init_column_matrix(int *matrix, size_t pitch)
{
uint xOffset = (blockIdx.x * blockDim.x) + threadIdx.x;
uint yOffset = (blockIdx.y * blockDim.y) + threadIdx.y;

uint skipX = gridDim.x * blockDim.x;
uint skipY = gridDim.y * blockDim.y;

while (xOffset < colCount)
{
while (yOffset < rowCount)
{
int *memoryRow = (int *)((char *)matrix + (xOffset * pitch));
memoryRow[yOffset] = (xOffset * rowCount) + yOffset;

yOffset += skipY;
}
xOffset += skipX;
}
}