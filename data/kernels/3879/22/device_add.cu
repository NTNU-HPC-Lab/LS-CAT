#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_add(int *a, int *b, int *c)
{
int blockId = blockIdx.x;
if (blockId < arrSize)
c[blockId] = a[blockId] + b[blockId];
}