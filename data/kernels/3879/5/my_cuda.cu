#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <functional>
#include <math.h>
#include <time.h>
#include <random>
#include <assert.h>


typedef unsigned char byte;

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
#define CUDA_CALL(err) (HandleError(err, __FILE__, __LINE__))

inline size_t get_number_of_parts(size_t whole, size_t divider)
{
    return ((whole + divider - 1) / divider);
}

#define CUDA_TIMED_BLOCK_START(fn_name)      \
    const char *___tmdFnName = fn_name;      \
    hipEvent_t startEvent, stopEvent;       \
    float elapsedTime;                       \
    CUDA_CALL(hipEventCreate(&startEvent)); \
    CUDA_CALL(hipEventCreate(&stopEvent));  \
    CUDA_CALL(hipEventRecord(startEvent, 0));

#define CUDA_TIMED_BLOCK_END                                              \
    CUDA_CALL(hipEventRecord(stopEvent, 0));                             \
    CUDA_CALL(hipEventSynchronize(stopEvent));                           \
    CUDA_CALL(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent)); \
    printf("%s took: %f ms\n", ___tmdFnName, elapsedTime);                \
    CUDA_CALL(hipEventDestroy(startEvent));                              \
    CUDA_CALL(hipEventDestroy(stopEvent));

template <typename T>
bool all_not_eq(const std::vector<T> &data, const T &cmp)
{
    for (size_t i = 0; i < data.size(); i++)
    {
        if (data[i] == cmp)
            return false;
    }
    return true;
}

template <typename T>
void safe_cuda_free(T *ptr)
{
    if (ptr != nullptr)
    {
        hipFree(ptr);
    }
}
