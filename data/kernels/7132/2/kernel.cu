#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* data, size_t from, size_t to, size_t min, size_t max, size_t NX)
{
size_t i = min + blockIdx.x * blockDim.x + threadIdx.x;
while (i < max) {
//TODO CONSIDER REMOVING MODULUS (might be slow)
if ( (i % NX != 0) && (i % NX != NX - 1) ){
data[to+i] = 0.2 * (
data[from+i]
+ data[from+i-1]
+ data[from+i+1]
+ data[from+i-NX]
+ data[from+i+NX]);
}
i +=gridDim.x*blockDim.x;
}
}