#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
int batch_index = blockIdx.x;
idx += m*nsample*batch_index;
grad_out += m*nsample*c*batch_index;
grad_points += n*c*batch_index;

int index = threadIdx.x;
int stride = blockDim.x;

for (int j=index;j<m;j+=stride) {
for (int k=0;k<nsample;++k) {
int ii = idx[j*nsample+k];
for (int l=0;l<c;++l) {
// Use atomic add to avoid race condition
atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
}
}
}
}