#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
int batch_index = blockIdx.x;
xyz1 += n*3*batch_index;
xyz2 += m*3*batch_index;
idx += m*nsample*batch_index;

int index = threadIdx.x;
int stride = blockDim.x;

for (int j=index;j<m;j+=stride) {
int cnt = 0;
for (int k=0;k<n;++k) {
if (cnt == nsample)
break; // only pick the FIRST nsample points in the ball
float x2=xyz2[j*3+0];
float y2=xyz2[j*3+1];
float z2=xyz2[j*3+2];
float x1=xyz1[k*3+0];
float y1=xyz1[k*3+1];
float z1=xyz1[k*3+2];
float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
if (d<radius) {
if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
for (int l=0;l<nsample;++l)
idx[j*nsample+l] = k;
}
idx[j*nsample+cnt] = k;
cnt+=1;
}
}
}
}