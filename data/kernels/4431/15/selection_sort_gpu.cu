#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
int batch_index = blockIdx.x;
dist+=m*n*batch_index;
outi+=m*n*batch_index;
out+=m*n*batch_index;

int index = threadIdx.x;
int stride = blockDim.x;

// copy from dist to dist_out
for (int j=index;j<m;j+=stride) {
for (int s=0;s<n;++s) {
out[j*n+s] = dist[j*n+s];
outi[j*n+s] = s;
}
}

float *p_dist;
for (int j=index;j<m;j+=stride) {
p_dist = out+j*n;
// selection sort for the first k elements
for (int s=0;s<k;++s) {
int min=s;
// find the min
for (int t=s+1;t<n;++t) {
if (p_dist[t]<p_dist[min]) {
min = t;
}
}
// swap min-th and i-th element
if (min!=s) {
float tmp = p_dist[min];
p_dist[min] = p_dist[s];
p_dist[s] = tmp;
int tmpi = outi[j*n+min];
outi[j*n+min] = outi[j*n+s];
outi[j*n+s] = tmpi;
}
}
}
}