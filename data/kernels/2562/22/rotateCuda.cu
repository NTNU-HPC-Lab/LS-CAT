#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotateCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle )
{
uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

// Only do something if this thread is for a valid pixel in the output
if ( outX < width && outY < height ) {
// Both input coordinates are shifted using the cosAngle, sinAngle, outX, and outY. The shift
// comes from inverse rotating the horizontal and vertical iterations over the output.

// Note that inverse rotation by X axis is [cos(angle), -sin(angle)],
//   and the inverse rotation by Y axis is [sin(angle),  cos(angle)].

const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

const int32_t inX = static_cast<int32_t>(exactInX);
const int32_t inY = static_cast<int32_t>(exactInY);

// Shift to the output pixel
out = out + outY * rowSizeOut + outX;

// Note that we will be taking an average with next pixels, so next pixels need to be in the image too
if ( inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1 ) {
*out = 0; // We do not actually know what is beyond the image, so set value to 0
}
else {
// Shift to the input pixel
in = in + inY * rowSizeIn + inX;

// Now we use a bilinear approximation to find the pixel intensity value. That is, we take an
// average of pixels (inX, inY), (inX + 1, inY), (inX, inY + 1), and (inX + 1, inY + 1).
// We add an offset of 0.5 so that conversion to integer is done using rounding.
const float probX = exactInX - inX;
const float probY = exactInY - inY;
const float mean = *in * (1 - probX) * (1 - probY) +
*(in + 1) * probX * (1 - probY) +
*(in + rowSizeIn) * (1 - probX) * probY +
*(in + rowSizeIn + 1) * probX * probY +
0.5f;

*out = static_cast<uint8_t>(mean);
}
}
}