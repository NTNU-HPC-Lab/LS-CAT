#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flipCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical )
{
const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

if ( inX < width && inY < height ) {
const uint32_t outX = horizontal ? (width  - 1 - inX) : inX;
const uint32_t outY = vertical   ? (height - 1 - inY) : inY;

out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
}
}