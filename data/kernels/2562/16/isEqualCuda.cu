#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isEqualCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t * isEqual )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
const uint32_t partsEqual = static_cast<uint32_t>( in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x] );
atomicAnd( isEqual, partsEqual );
}
}