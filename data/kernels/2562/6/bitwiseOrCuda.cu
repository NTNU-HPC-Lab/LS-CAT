#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitwiseOrCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
const uint32_t idIn1 = y * rowSizeIn1 + x;
const uint32_t idIn2 = y * rowSizeIn2 + x;
const uint32_t idOut = y * rowSizeOut + x;
out[idOut] = in1[idIn1] | in2[idIn2];
}
}