#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void revisedArraySum(float *array, float *sum){

__shared__ float partialSum[256];
int t = threadIdx.x;
for(int stride = 1;stride < blockDim.x; stride *= 2){
__syncthreads();
if(t % (2 * stride) == 0){
partialSum[t] += partialSum[t + stride];
}
}
sum[0] = partialSum[0];
}