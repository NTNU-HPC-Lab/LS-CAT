#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_min_gpu(float *device_input, float *device_output){
extern __shared__ float sm[];

int tid = threadIdx.x;
int i = blockIdx.x * blockDim.x + threadIdx.x;

sm[tid] = device_input[i];
__syncthreads();

for(int s = 1;s < blockDim.x; s*= 2){
if(tid % (2 * s) == 0){
sm[tid] = min(sm[tid], sm[tid+s]);
}
__syncthreads();
}

if(tid == 0) device_output[blockIdx.x] = sm[0];
}