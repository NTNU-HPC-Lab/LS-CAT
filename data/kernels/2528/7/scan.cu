#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float *input, float *output, int len) {
__shared__ float data[BLOCK_SIZE];

// DEBUG
if (threadIdx.x == 0 && blockIdx.x == 0)
{
printf("Block Number: %d\n", blockIdx.x);
for (int i = 0; i < BLOCK_SIZE; ++i)
{
printf("DATA[%d] = %f\n", i, data[i]);
}
}

}