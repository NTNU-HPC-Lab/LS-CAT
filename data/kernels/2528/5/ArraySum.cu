#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ArraySum(float *array, float *sum){
int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < N){
atomicAdd(sum, array[index]);
}
}