#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAddGPU(double *a, double *b, double *c, double n){

int id = blockIdx.x * blockDim.x + threadIdx.x;

if (id < n){
c[id] = a[id] + b[id];
}
}