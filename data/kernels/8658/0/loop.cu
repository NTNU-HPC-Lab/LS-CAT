#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Refactor `loop` to be a CUDA Kernel. The new kernel should
* only do the work of 1 iteration of the original loop.
*/


__global__ void loop()
{
printf("This is iteration number %d\n", threadIdx.x);
}