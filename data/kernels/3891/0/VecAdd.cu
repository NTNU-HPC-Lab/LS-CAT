#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(int n, const float *A, const float *B, float* C) {
//DEVICE(GPU)CODE
/********************************************************************
*
* Compute C = A + B
*   where A is a (1 * n) vector
*   where B is a (1 * n) vector
*   where C is a (1 * n) vector
*
********************************************************************/
//added for extra compute time
long long start = clock64();
long long cycles_elapsed;
do{cycles_elapsed = clock64() - start;}
while(cycles_elapsed <20000);
//end of added compute time
// INSERT KERNEL CODE HERE
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < n)
C[i] = A[i] + B[i];
}