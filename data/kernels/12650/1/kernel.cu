#include "hip/hip_runtime.h"
#include "includes.h"


#define FIBER 32
#define MATRIX_SIZE 2048
#define DATA_SIZE MATRIX_SIZE * MATRIX_SIZE * sizeof(int)
#define MAX_MATRIX_SIZE (MATRIX_SIZE * MATRIX_SIZE)



using namespace std;

__global__ void kernel(int *A, int *C, int *B, int *result) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int first_index = i + j * MATRIX_SIZE;
int second_index = j + i * MATRIX_SIZE;

if (first_index < MAX_MATRIX_SIZE && second_index < MAX_MATRIX_SIZE)
{
result[first_index] = (A[first_index] + A[first_index]) * B[second_index] - C[first_index];
}
}