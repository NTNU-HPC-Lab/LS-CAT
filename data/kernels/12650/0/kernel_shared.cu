#include "hip/hip_runtime.h"
#include "includes.h"


#define FIBER 32
#define MATRIX_SIZE 2048
#define DATA_SIZE MATRIX_SIZE * MATRIX_SIZE * sizeof(int)
#define MAX_MATRIX_SIZE (MATRIX_SIZE * MATRIX_SIZE)



using namespace std;

__global__ void kernel_shared(int *A, int *C, int *B, int *result) {
__shared__ int shared_memory[FIBER][FIBER];

int i = blockIdx.x * blockDim.x + threadIdx.y;
int j = blockIdx.y * blockDim.y + threadIdx.x;

shared_memory[threadIdx.y][threadIdx.x] = B[i * MATRIX_SIZE + j];

__syncthreads();

i = blockIdx.x * blockDim.x + threadIdx.x;
j = blockIdx.y * blockDim.y + threadIdx.y;

int first_index = i + j * MATRIX_SIZE;
int second_index = j + i * MATRIX_SIZE;

if (first_index < MAX_MATRIX_SIZE && second_index < MAX_MATRIX_SIZE)
{
result[first_index] = (A[first_index] + A[first_index]) * shared_memory[threadIdx.x][threadIdx.y] - C[first_index];
}
}