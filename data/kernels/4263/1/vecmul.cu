#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecmul(float *A, float* B, float *C, int size)
{
// Row and Column indexes:
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

// Are they bellow the maximum?
if (col < size && row < size) {
float result = 0;
for(int ix=0;ix<size;ix++) {
result += A[row*size+ix]*B[ix*size+col];
}
C[row*size+col] = result;
}
}