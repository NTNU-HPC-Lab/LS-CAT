#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hello_from_gpu()
{
const int b = blockIdx.x;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
printf("Hello World from block-%d and thread-(%d, %d)!\n", b, tx, ty);
}