#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readGlobalMemoryUnit(float *data, float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
float sum = 0;
int s = gid*512;
for (j=0 ; j<repeat ; ++j)
{
float a0 = data[(s+0)&(size-1)];
float a1 = data[(s+1)&(size-1)];
float a2 = data[(s+2)&(size-1)];
float a3 = data[(s+3)&(size-1)];
float a4 = data[(s+4)&(size-1)];
float a5 = data[(s+5)&(size-1)];
float a6 = data[(s+6)&(size-1)];
float a7 = data[(s+7)&(size-1)];
float a8 = data[(s+8)&(size-1)];
float a9 = data[(s+9)&(size-1)];
float a10 = data[(s+10)&(size-1)];
float a11 = data[(s+11)&(size-1)];
float a12 = data[(s+12)&(size-1)];
float a13 = data[(s+13)&(size-1)];
float a14 = data[(s+14)&(size-1)];
float a15 = data[(s+15)&(size-1)];
sum += a0+a1+a2+a3+a4+a5+a6+a7+a8+a9+a10+a11+a12+a13+a14+a15;
s = (s+16)&(size-1);
}
output[gid] = sum;
}