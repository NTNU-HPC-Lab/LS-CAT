#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop3b(float *delta_nabla_b,int b_off,float *zs) {
delta_nabla_b[b_off+threadIdx.x]*=(1.0/(1.0+expf(-zs[threadIdx.x])))*(1.0-(1.0/(1.0+expf(-zs[threadIdx.x]))));
}