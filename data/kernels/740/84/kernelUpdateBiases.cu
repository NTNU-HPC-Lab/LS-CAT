#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateBiases(float *nabla_b,float *biases,float eta,float mini_batch_size) {

float rate=eta/mini_batch_size;

biases[threadIdx.x]-=rate*nabla_b[threadIdx.x];
}