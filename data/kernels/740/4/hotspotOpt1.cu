#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc, int nx, int ny, int nz, float ce, float cw, float cn, float cs, float ct, float cb, float cc)
{
float amb_temp = 80.0;

int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

int c = i + j * nx;
int xy = nx * ny;

int W = (i == 0)        ? c : c - 1;
int E = (i == nx-1)     ? c : c + 1;
int N = (j == 0)        ? c : c - nx;
int S = (j == ny-1)     ? c : c + nx;

float temp1, temp2, temp3;
temp1 = temp2 = tIn[c];
temp3 = tIn[c+xy];
tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
+ cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
c += xy;
W += xy;
E += xy;
N += xy;
S += xy;

for (int k = 1; k < nz-1; ++k) {
temp1 = temp2;
temp2 = temp3;
temp3 = tIn[c+xy];
tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
+ cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
c += xy;
W += xy;
E += xy;
N += xy;
S += xy;
}
temp1 = temp2;
temp2 = temp3;
tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
+ cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
return;
}