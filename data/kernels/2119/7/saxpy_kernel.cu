#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_kernel(int n, float a, float *v1, float *v2, float *s){
int i = blockIdx.x*blockDim.x + threadIdx.x;
if ( i < n ) s[i] = a*v1[i] + v2[i];
}