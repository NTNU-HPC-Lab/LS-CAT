#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_saxpy(int n, float a, float *x, float *y, float *s)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < n) s[i] = a*x[i] + y[i];
}