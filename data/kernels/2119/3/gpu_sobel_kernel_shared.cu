#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_sobel_kernel_shared(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
__shared__ u_char tuile[BLOCKDIM_X][BLOCKDIM_Y];

int x = threadIdx.x;
int y = threadIdx.y;
int i = blockIdx.y*(BLOCKDIM_Y-2) + y;
int j = blockIdx.x*(BLOCKDIM_X-2) + x;

int globalIndex = i*width+j;

if ((i==0)||(i>=height-1)||(j==0)||(j>=width-1)) {}
else {
//mainstream
tuile[x][y] = Source[globalIndex];
__syncthreads();

u_char val;
if ((x>0)&&(y>0)&&(x<BLOCKDIM_X-1)&&(y<BLOCKDIM_Y-1)) {
val = std::abs(tuile[x-1][y-1] + tuile[x-1][y] + tuile[x-1][y+1] -\
(tuile[x+1][y-1] + tuile[x+1][y] + tuile[x+1][y+1]));
Resultat[globalIndex]  = val + std::abs(tuile[x-1][y-1] + tuile[x][y-1] + tuile[x+1][y-1] -\
(tuile[x-1][y+1] + tuile[x][y+1] + tuile[x+1][y+1]));
}
}
}