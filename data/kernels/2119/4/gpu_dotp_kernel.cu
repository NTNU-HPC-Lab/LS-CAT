#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_dotp_kernel(int size, float* vec1, float* vec2, float* res){

float cache = 0;
int i = blockIdx.x * blockDim.x + threadIdx.x;

if( i < size ){
cache = vec1[i]*vec2[i];
}

atomicAdd(res, cache);
}