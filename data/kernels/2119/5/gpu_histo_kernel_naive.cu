#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_histo_kernel_naive(u_char* Source, int *res, unsigned height, unsigned width){
int j = blockIdx.x*blockDim.x + threadIdx.x;
int i = blockIdx.y*blockDim.y + threadIdx.y;
if ((i<0)||(i>=height)||(j<0)||(j>=width)) {}
else {
u_char val = Source[i*width+j];
atomicAdd(&res[val],1);
}
}