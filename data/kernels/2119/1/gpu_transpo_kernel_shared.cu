#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpo_kernel_shared(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
__shared__ u_char tuile[BLOCKDIM_X][BLOCKDIM_Y+1];

int x = threadIdx.x;
int y = threadIdx.y;
int i = blockIdx.y*(BLOCKDIM_Y) + y;
int j = blockIdx.x*(BLOCKDIM_X) + x;


if ((i<0)||(i>=height)||(j<0)||(j>=width)) {}
else {
tuile[y][x] = Source[i*width + j];
__syncthreads();
int i = blockIdx.y*(BLOCKDIM_Y) + x;
int j = blockIdx.x*(BLOCKDIM_X) + y;
Resultat[j*height + i] = tuile[x][y];
}
}