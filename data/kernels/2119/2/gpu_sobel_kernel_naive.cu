#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_sobel_kernel_naive(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
int j = blockIdx.x*blockDim.x + threadIdx.x;
int i = blockIdx.y*blockDim.y + threadIdx.y;
u_char val;
int globalIndex = i*width+j;
if ((i==0)||(i>=height-1)||(j==0)||(j>=width-1)) {Resultat[globalIndex]=0;}
else {
val  = std::abs(Source[(i-1)*width+(j-1)] + Source[(i-1)*width+(j)] + Source[(i-1)*width+(j+1)] -\
(Source[(i+1)*width+(j-1)] + Source[(i+1)*width+(j)] + Source[(i+1)*width+(j+1)]));
Resultat[globalIndex]  = val + std::abs(Source[(i-1)*width+(j-1)] + Source[(i)*width+(j-1)] + Source[(i+1)*width+(j-1)] -\
(Source[(i-1)*width+(j+1)] + Source[(i)*width+(j+1)] + Source[(i+1)*width+(j+1)]));

}
}