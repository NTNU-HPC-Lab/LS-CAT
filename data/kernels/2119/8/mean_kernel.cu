#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(int n, float* v1, float* v2, float* res){
int i = threadIdx.x + blockIdx.x*blockDim.x;
if( i < n ) res[i] = (v1[i] + v2[i])/2;
}