#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpo_kernel_naive(u_char *Source, u_char *Resultat, unsigned width, unsigned height){
int j = blockIdx.x*blockDim.x + threadIdx.x;
int i = blockIdx.y*blockDim.y + threadIdx.y;

if ((i<0)||(i>=height)||(j<0)||(j>=width)) {}
else {
Resultat[j*height + i]  = Source[i*width + j];
}
}