#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PD_ZC_GPU_KERNEL(float *d_input, float *d_output, int maxTaps, int nTimesamples, int nLoops)
{
int x_r, y_r, x_w, y_w;
int Elements_per_block = PD_NTHREADS * PD_NWINDOWS;

//read
y_r = ( blockIdx.y * blockDim.y + threadIdx.y ) * nTimesamples;
x_r = ( blockIdx.x + 1 ) * Elements_per_block + threadIdx.x;

//write
y_w = ( blockIdx.y * blockDim.y + threadIdx.y ) * ( maxTaps - 1 ) * gridDim.x;
x_w = blockIdx.x * ( maxTaps - 1 ) + threadIdx.x;

for (int f = 0; f < nLoops; f++)
{
if (x_r < nTimesamples && threadIdx.x < ( maxTaps - 1 ))
{
d_output[x_w + y_w + f * WARP] = d_input[x_r + y_r + f * WARP];
}
}
}