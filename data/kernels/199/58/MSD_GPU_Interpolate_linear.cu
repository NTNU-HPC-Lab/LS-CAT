#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MSD_GPU_Interpolate_linear(float *d_MSD_DIT, float *d_MSD_interpolated, int *d_MSD_DIT_widths, int MSD_DIT_size, int *boxcar, int max_width_performed){

int tid  = threadIdx.x;
if(boxcar[tid] <= max_width_performed) {
//      int f = threadIdx.x;
int desired_width = boxcar[tid];
int position = (int) floorf(log2f((float) desired_width));

float width1 = d_MSD_DIT_widths[position];
float mean1 = d_MSD_DIT[(position)*MSD_RESULTS_SIZE];
float StDev1 = d_MSD_DIT[(position)*MSD_RESULTS_SIZE +1];

//      printf("\nBoxcar: %f \t desired: %f", (float)boxcar[f], desired_width);

if(position == MSD_DIT_size-1 && width1==(int) desired_width) {
//                (*mean) = mean1;
//                (*StDev) = StDev1;
d_MSD_interpolated[tid*2] = mean1;
d_MSD_interpolated[tid*2+1] = StDev1;
}
else {
float width2 = d_MSD_DIT_widths[position+1];
float distance_in_width = width2 - width1;

float mean2 = d_MSD_DIT[(position+1)*MSD_RESULTS_SIZE];
float distance_in_mean = mean2 - mean1;

float StDev2 = d_MSD_DIT[(position+1)*MSD_RESULTS_SIZE +1];
float distance_in_StDev = StDev2 - StDev1;

//                        printf("Position: \t %i \t f: %i\n", position, f);
//                        printf("width:[%f;%f]; mean:[%f;%f]; sd:[%f;%f]\n",width1, width2, mean1, mean2, StDev1, StDev2);
//                        printf("d width %f; d mean: %f; d StDef: %f\n", distance_in_width, distance_in_mean, distance_in_StDev);
//                        printf("\tDesired_width: %f\n", desired_width);

//                (*mean) = mean1 + (distance_in_mean/distance_in_width)*((float) desired_width - width1);
//                (*StDev) = StDev1 + (distance_in_StDev/distance_in_width)*((float) desired_width - width1);
d_MSD_interpolated[tid*2] = mean1 + (distance_in_mean/distance_in_width)*((float) desired_width - width1);
d_MSD_interpolated[tid*2+1] = StDev1 + (distance_in_StDev/distance_in_width)*((float) desired_width - width1);

}
}
}