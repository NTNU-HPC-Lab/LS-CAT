#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

size_t t = blockIdx.x * blockDim.x + threadIdx.x;
size_t c = blockIdx.y * blockDim.y + threadIdx.y;

d_input[(size_t)(c * nsamp) + t] = (unsigned short) __ldg(&d_output[(size_t)(c * nsamp) + t]);

}