#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bootstrap3(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
float myResample;

int constant = ( 4294967295 / ( bins ) );
int id = threadIdx.x + blockDim.x * blockIdx.x;
int dmid = bins * ( blockDim.y * blockIdx.y + threadIdx.y );
for (int i = 0; i < bins; i++)
{

int rid = g_irand[id * bins + i] / constant;

myResample += g_idata[dmid + rid];
}
dmid = num_boots * ( blockDim.y * blockIdx.y + threadIdx.y );
g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ( (double) myResample / (double) num_els );

}