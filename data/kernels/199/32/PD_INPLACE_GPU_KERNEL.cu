#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PD_INPLACE_GPU_KERNEL(float *d_input, float *d_temp, unsigned char *d_output_taps, float *d_MSD, int maxTaps, int nTimesamples)
{
extern __shared__ float s_input[]; //dynamically allocated memory for now

int f, i, gpos_y, gpos_x, spos, itemp;
float res_SNR[PD_NWINDOWS], SNR, temp_FIR_value, FIR_value, ftemp;
int res_Taps[PD_NWINDOWS];
float signal_mean, signal_sd, modifier;
signal_mean = d_MSD[0];
signal_sd = d_MSD[2];
modifier = d_MSD[1];

//----------------------------------------------
//----> Reading data
gpos_y = blockIdx.y * nTimesamples;
gpos_x = blockIdx.x * PD_NTHREADS * PD_NWINDOWS + threadIdx.x;
spos = threadIdx.x;
for (f = 0; f < PD_NWINDOWS; f++)
{
if (gpos_x < nTimesamples)
{
s_input[spos] = d_input[gpos_y + gpos_x];
}
spos = spos + blockDim.x;
gpos_x = gpos_x + blockDim.x;
}

//----> Loading shared data
itemp = PD_NTHREADS * PD_NWINDOWS + maxTaps - 1;
gpos_y = blockIdx.y * ( maxTaps - 1 ) * gridDim.x;
gpos_x = blockIdx.x * ( maxTaps - 1 ) + threadIdx.x;
while (spos < itemp)
{ // && gpos_x<((maxTaps-1)*gridDim.x)
s_input[spos] = d_temp[gpos_y + gpos_x];
spos = spos + blockDim.x;
gpos_x = gpos_x + blockDim.x;
}

__syncthreads();

//----> SNR for nTaps=1
spos = PD_NWINDOWS * threadIdx.x;
for (i = 0; i < PD_NWINDOWS; i++)
{
res_SNR[i] = ( s_input[spos + i] - signal_mean ) / signal_sd;
res_Taps[i] = 1;
}

//----------------------------------------------
//----> FIR calculation loop
FIR_value = s_input[spos];
for (f = 1; f < maxTaps; f++)
{
//nTaps=f+1;!
ftemp = signal_sd + f * modifier;
spos = PD_NWINDOWS * threadIdx.x;

// 0th element from NWINDOW
i = 0;
FIR_value += s_input[spos + f];

SNR = ( FIR_value - ( f + 1 ) * signal_mean ) / ( ftemp );
if (SNR > res_SNR[i])
{
res_SNR[i] = SNR;
res_Taps[i] = f + 1;
}

temp_FIR_value = FIR_value;
for (i = 1; i < PD_NWINDOWS; i++)
{
temp_FIR_value = temp_FIR_value - s_input[spos + i - 1] + s_input[spos + f + i];

SNR = ( temp_FIR_value - ( f + 1 ) * signal_mean ) / ( ftemp );
if (SNR > res_SNR[i])
{
res_SNR[i] = SNR;
res_Taps[i] = f + 1;
}
}
}

//----------------------------------------------
//---- Writing data
gpos_y = blockIdx.y * nTimesamples;
gpos_x = blockIdx.x * PD_NTHREADS * PD_NWINDOWS + PD_NWINDOWS * threadIdx.x;
for (i = 0; i < PD_NWINDOWS; i++)
{
if (( gpos_x + i ) < ( nTimesamples ))
{
d_input[gpos_y + gpos_x + i] = res_SNR[i];
d_output_taps[gpos_y + gpos_x + i] = res_Taps[i];
}
}
}