#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bootstrap2(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
float myResample = 0.0f;

int constant = ( 4294967295 / ( bins ) );
int constant2 = blockIdx.x * bins;
int dmid = bins * ( blockDim.y * blockIdx.y + threadIdx.y );
for (int i = 0; i < bins; i++)
{

int rid = g_irand[constant2 + i] / constant;

myResample += g_idata[dmid + ( ( rid + threadIdx.x ) % bins )];
}
dmid = num_boots * ( blockDim.y * blockIdx.y + threadIdx.y );
g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ( (double) myResample / (double) num_els );
}