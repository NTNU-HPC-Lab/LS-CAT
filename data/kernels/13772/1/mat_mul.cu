#include "hip/hip_runtime.h"
#include "includes.h"
//Source: https://kb.iu.edu/d/bdmg
//INDIANA UNIVERSITY
/**********************    mat_mul.cu    ******************************/

#define M  256
#define P  128
#define N   64
#define BLKSIZ 16


/**********************************************************************/
__global__ void mat_mul(float *Ad, float *Bd, float *Cd) {
int    m = blockIdx.x;
int    n = blockIdx.y;
int    i = threadIdx.x;
int    j = threadIdx.y;
int    k,p;
float  c = 0.0;

__shared__  float As[BLKSIZ][BLKSIZ];
__shared__  float Bs[BLKSIZ][BLKSIZ];

for(p=0;p<P/BLKSIZ;p++) {
As[i][j] = Ad[(m*BLKSIZ+i)*P+(p*BLKSIZ+j)];
Bs[i][j] = Bd[(p*BLKSIZ+i)*N+(n*BLKSIZ+j)];
__syncthreads();
for(k=0; k<BLKSIZ; k++) {
c += As[i][k] * Bs[k][j];
}
}
Cd[(m*BLKSIZ+i)*N+(n*BLKSIZ+j)] = c;
}