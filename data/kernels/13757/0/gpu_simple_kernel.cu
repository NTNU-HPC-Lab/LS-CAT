#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_simple_kernel(float* a, float* b, float* c, int N)
{
//int thread_idx = threadIdx.x;

int idx = blockIdx.x*blockDim.x + threadIdx.x;
if ( idx > N)
return;

#define PRINT_IDS
#if !defined( __CUDA_ARCH__) || (__CUDA_ARCH__ >= 200 ) &&  defined(PRINT_IDS)
// Check nvcc compiler gencode
// at least -gencode=arch=compute_20,code=\"sm_20,compute_20\" should be set
printf("thread: %3d - block: %3d - threadIdx: %3d, warp: %3d\n", idx, blockIdx.x, threadIdx.x, threadIdx.x/warpSize );
#endif

c[idx] = a[idx] * b[idx];
}