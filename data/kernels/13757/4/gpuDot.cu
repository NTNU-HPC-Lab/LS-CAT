#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuDot(float* dot, float* a, float* b, int N)
{
__shared__ float cache[THREADS_PER_BLOCK];
int tid = blockIdx.x*blockDim.x + threadIdx.x;
int cacheIdx = threadIdx.x;

float temp = 0;

while (tid < N)
{
temp += a[tid] * b[tid];
tid += blockDim.x * gridDim.x;
}

cache[cacheIdx]=temp;

__syncthreads();

int i = blockDim.x/2;
while (i != 0)
{
if (cacheIdx < i)
cache[cacheIdx] += cache[cacheIdx + i];

__syncthreads();
i /= 2;
}

if (cacheIdx == 0)
dot[blockIdx.x] = cache[0];
}