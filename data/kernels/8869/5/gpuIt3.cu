#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuIt3(float *tNew,float *tOld,float *tOrig,int x,int y,int z,float k,float st) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
if(i < x*y*z){

if(i == 0){ // front upper left corner
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 0;
}
else if(i == x-1){ // front upper right corner
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = .1;
}
else if(i == x*y-1){ // front lower right corner
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = .2;
}
else if(i == x*y-x){ // front lower left corner
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = .3;
}
else if(i == x*y*(z-1) ){ // back upper left corner
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = .4;
}
else if(i == x*y*(z-1) + x-1){ // back upper right corner
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = .5;
}
else if(i == x*y*z-1){ // back lower right corner
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = .6;
}
else if(i == x*y*z - x){ // back lower left corner
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = .7;
}

else if(i - x < 0){ // front top edge
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = .8;
}
else if(i%x == x-1 && i<x*y){ // front right edge
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = .9;
}
else if(i+x > x*y && i < (x*y)){ // front bottom edge
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1;
}
else if(i%x == 0 && i<x*y){ // front left edge
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 2;
}

else if(i > (x*y*z - x*y) && i < (x*y*z - (x-1)*y)){ // back top edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 3;
}
else if(i%x == x-1 && i > (x*y*(z-1))){ // back right edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = 4;
}
else if(i+x > x*y*z){ // back bottom edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 5;
}
else if(i%x == 0 && i > x*y*(z-1)){ // back left edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 6;
}

// the corner sides going front to back
else if(i%(x*y) == 0){ // upper left edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 7;
}
else if(i%(x*y) == x-1){ // upper right edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = 8;
}
else if(i%(x*y) == x*y-1){ // lower right edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = 9;
}
else if(i%(x*y) == x*y-x){ // lower left edge
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 9.1;
}

// else ifs here are vague because other options already completed
else if(i < x*y){ // front face
tNew[i] = tOld[i]+k*(tOld[i]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1.1;
}
else if(i > x*y*(z-1)){ // back face
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1.2;
}
else if(i%(x*y) < x){ // top face
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1.3;
}
else if(i%(x*y) > x*(y-1)){ // bottom face
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1.4;
}
else if(i%(x) == x-1){ // right face
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i]-6*tOld[i]);
//tNew[i] = 1.5;
}
else if(i%(x) == 0){ // left face
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i]+tOld[i+1]-6*tOld[i]);
//tNew[i] = 1.6;
}
else{ // all in the middle
//                       front        back         top       bottom     left     right
tNew[i] = tOld[i]+k*(tOld[i-(x*y)]+tOld[i+(x*y)]+tOld[i-x]+tOld[i+x]+tOld[i-1]+tOld[i+1]-6*tOld[i]);
}


//tNew[i] = i%(x*y);
// replace heaters
if(tOrig[i] != st){
tNew[i] = tOrig[i];
}

}
}