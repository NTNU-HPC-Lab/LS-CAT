#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuIt(float *tNew,float *tOld,float *tOrig,int x,int y,int z,float k,float st) {

int i = threadIdx.x + blockIdx.x * blockDim.x;
// may want an if(i < x*y*z) to prevent overflowing, likea thisa
if(i < x*y*z){

if(i == 0){ // top left corner
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i] + tOld[i] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 1;
}
else if(i == x-1){ // top right corner
tNew[i] = tOld[i] + k*(tOld[i] + tOld[i-1] + tOld[i] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 3;
}
else if(i == x*y - 1){ // bottom right corner
tNew[i] = tOld[i] + k*(tOld[i] + tOld[i-1] + tOld[i-x] + tOld[i] - 4*tOld[i]);
//tNew[i] = 5;
}
else if(i == x*y - x){ // bottom left corner
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i] + tOld[i-x] + tOld[i] - 4*tOld[i]);
//tNew[i] = 7;
}
else if(i%x == 0){ // left side
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i] + tOld[i-x] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 8;
}
else if(i%x == x-1){ // right side
tNew[i] = tOld[i] + k*(tOld[i] + tOld[i-1] + tOld[i-x] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 4;
}
else if(i - x < 0){ // top row
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i-1] + tOld[i] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 2;
}
else if(i + x > x*y){ // bottom row
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i-1] + tOld[i-x] + tOld[i] - 4*tOld[i]);
//tNew[i] = 6;
}
else{
tNew[i] = tOld[i] + k*(tOld[i+1] + tOld[i-1] + tOld[i-x] + tOld[i+x] - 4*tOld[i]);
//tNew[i] = 9;
}
//tNew[i] = i; // for debugging
// replace heaters
if(tOrig[i] != st){
tNew[i] = tOrig[i];
}
//tNew[i] = i%x;
}
}