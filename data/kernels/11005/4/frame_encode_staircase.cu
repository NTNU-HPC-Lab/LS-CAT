#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void frame_encode_staircase(int *data, int *pcm,int param_k,int param_m,int w_f,int packet_size)
{
int ps = packet_size/sizeof(int);

int x  = threadIdx.x;

for (int index = param_k; index < param_k + param_m-1; index++)
{

int offset = x;

while (offset < ps)
{
// *((int *)(data + (index+1)*ps + offset + intSize * x)) ^= *((int *)(data + index * ps + intSize * x + offset));
data[(index+1)*ps + offset] ^= data[index*ps + offset];
offset += blockDim.x;
}



}


}