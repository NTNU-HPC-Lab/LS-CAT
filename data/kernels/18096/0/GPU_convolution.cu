#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32

struct event_pair
{
hipEvent_t start;
hipEvent_t end;
};

__global__ void GPU_convolution(float *channel, float *mask, float *result, int dimMask, int dimW, int dimH) {
int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int x, y;

// Identify the row and column of the Pd element to work on
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;

int nidRow = Row - dimMask / 2;
int nidCol = Col - dimMask / 2;

int tid = Row * dimW + Col;

if (tid < dimW * dimH) {
result[tid] = 0;
for (int i = 0; i < dimMask; ++i) {
x = nidRow * dimW + i * dimW;
for (int j = 0; j < dimMask; ++j) {
y = nidCol + j;
// When the value is not beyond the borders
if (x >= 0 && y >= 0 && x < dimW * dimH && y < dimW) {
result[tid] += mask[dimMask * i + j] * channel[x + y];
}
}
}
if (result[tid] > 255)
result[tid] = 255;
if (result[tid] < 0)
result[tid] = 0;
}
}