#include "hip/hip_runtime.h"
#include "includes.h"



__device__ float fitness_function(float x[])
{
float y,yp;
float res=0;
float y1=1+(x[0]-1)/4;
float yn=1+(x[NUM_OF_DIMENSIONS-1]-1)/4;

res+=pow(sin(phi*y1),2)+pow(yn-1,2);

for(int i=0;i<NUM_OF_DIMENSIONS-1;i++)
{
y=1+(x[i]-1)/4;
yp=1+(x[i+1]-1)/4;
res+=pow(y-1,2)*(1+10*pow(sin(phi*yp),2));
}

return res;
}
__global__ void kernelUpdateGBest(float *gBest,float *pBests)
{
float temp[NUM_OF_DIMENSIONS];
for(int i=0;i<10*NUM_OF_DIMENSIONS;i+=NUM_OF_DIMENSIONS)
{
for(int k=0;k<NUM_OF_DIMENSIONS;k++)
temp[k]=pBests[i+k];

if(fitness_function(temp)<fitness_function(gBest))
{
for(int k=0;k<NUM_OF_DIMENSIONS;k++)
gBest[k]=temp[k];
}
}
}