#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void kernelUpdateParticle(float *positions,float *velocities,float *pBests,float *gBest,float r1,float r2)
{
int i=blockIdx.x*blockDim.x+threadIdx.x;
if(i>=NUM_OF_PARTICLES*NUM_OF_DIMENSIONS)
return;

float rp=r1;
float rg=r2;

velocities[i]=OMEGA*velocities[i]+c1*rp*(pBests[i]-positions[i])+c2*rg*(gBest[i%NUM_OF_DIMENSIONS]-positions[i]);
positions[i]+=velocities[i];
}