#include "hip/hip_runtime.h"
#include "includes.h"



__device__ float fitness_function(float x[])
{
float y,yp;
float res=0;
float y1=1+(x[0]-1)/4;
float yn=1+(x[NUM_OF_DIMENSIONS-1]-1)/4;

res+=pow(sin(phi*y1),2)+pow(yn-1,2);

for(int i=0;i<NUM_OF_DIMENSIONS-1;i++)
{
y=1+(x[i]-1)/4;
yp=1+(x[i+1]-1)/4;
res+=pow(y-1,2)*(1+10*pow(sin(phi*yp),2));
}

return res;
}
__global__ void kernelUpdatePBest(float *positions,float *pBests,float *gBest)
{
int i=blockIdx.x*blockDim.x+threadIdx.x;
if(i>=NUM_OF_PARTICLES*NUM_OF_DIMENSIONS||i%NUM_OF_DIMENSIONS!=0)
return;

float tempParticle1[NUM_OF_DIMENSIONS];
float tempParticle2[NUM_OF_DIMENSIONS];

for(int j=0;j<NUM_OF_DIMENSIONS;j++)
{
tempParticle1[j]=positions[i+j];
tempParticle2[j]=pBests[i+j];
}

if(fitness_function(tempParticle1)<fitness_function(tempParticle2))
{
for(int j=0;j<NUM_OF_DIMENSIONS;j++)
pBests[i+j]=tempParticle1[j];

if(fitness_function(tempParticle1)<fitness_function(gBest))
{
for(int j=0;j<NUM_OF_DIMENSIONS;j++)
atomicExch(gBest+j,tempParticle1[j]);
}
}
}