#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

int *a, *b;  // host data
int *c, *c2;  // results


__global__ void vecAdd(int *A,int *B,int *C,int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
C[i] = A[i] + B[i];
}