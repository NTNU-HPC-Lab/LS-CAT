#include "hip/hip_runtime.h"
#include "includes.h"






// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

//utilizacion del teorema de pitagoras a lo largo del vector en el gpu

//creacion de un buffer de imagenes, regresando al host, pasando del dispositivo al host de puntero a puntero
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, size_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
//Distribucion de indices para la localizacion de los pixeles
int x = (blockIdx.x * blockDim.x) + threadIdx.x;
int y = (blockIdx.y * blockDim.y) + threadIdx.y;

float sum = 0.0;
int   pWidth = kWidth / 2;
int   pHeight = kHeight / 2;
//Solo ejecutamos para pixeles validos
if (x >= pWidth + paddingX &&
y >= pHeight + paddingY &&
x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
y < (blockDim.y * gridDim.y) - pHeight - paddingY)
{
for (int j = -pHeight; j <= pHeight; j++)
{
for (int i = -pWidth; i <= pWidth; i++)
{
//obteniendo el peso para la locacion
int ki = (i + pWidth);
int kj = (j + pHeight);
float w = convolutionKernelStore[(kj * kWidth) + ki + kOffset];
sum += w * float(source[((y + j) * width) + (x + i)]);
}
}
}
//Promedio de la suma
destination[(y * width) + x] = (unsigned char)sum;
}