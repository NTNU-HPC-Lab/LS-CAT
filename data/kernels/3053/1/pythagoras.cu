#include "hip/hip_runtime.h"
#include "includes.h"






// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

//utilizacion del teorema de pitagoras a lo largo del vector en el gpu

//creacion de un buffer de imagenes, regresando al host, pasando del dispositivo al host de puntero a puntero
__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

float af = float(a[idx]);
float bf = float(b[idx]);

c[idx] = (unsigned char)sqrtf(af*af + bf * bf);
}