#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_long(long* x, int leng) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < leng) {
printf("%ld,", x[ i ]);
}
}