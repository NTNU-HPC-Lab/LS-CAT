#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_float(float* x, int leng) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < leng) {
printf("%f,", x[ i ]);
}
}