#include "hip/hip_runtime.h"
#include "includes.h"


__global__ static void mapToGLKernel(uint8_t *dSrc, uint8_t *dDst, int nWidth, int nHeight) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
if (tid < nWidth && tidd < nHeight) {
int j = tidd * nWidth * 3;
int k = tid * 3;
dDst[j + k + 0] = dSrc[j + k + 0];
dDst[j + k + 1] = dSrc[j + k + 1];
dDst[j + k + 2] = dSrc[j + k + 2];
}
}