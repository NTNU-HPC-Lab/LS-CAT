#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *dpDst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int tidd = blockIdx.y * blockDim.y + threadIdx.y;
uint4 pF;
int scale = nSrcHeight / nDstHeight;
if (scale == 4) {
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 6;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 32;

pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
pF.w = (uint32_t)p_Src[j + k + 6];

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);
y1 = (uint32_t)(pF.w & 0x000003FF);

pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
pF.z = (uint32_t)p_Src[j + k + 12];

y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
v1 = (uint32_t)(pF.z & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
y4 = (uint32_t)(pF.w & 0x000003FF);

pF.y = ((uint32_t)p_Src[j + k + 27] << 16);

y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

k = tid * 6;
j = tidd * nDstWidth;
dpDst[j + k + 0] = lookupTable_cuda[y0];
dpDst[j + k + 1] = lookupTable_cuda[y1];
dpDst[j + k + 2] = lookupTable_cuda[y2];
dpDst[j + k + 3] = lookupTable_cuda[y3];
dpDst[j + k + 4] = lookupTable_cuda[y4];
dpDst[j + k + 5] = lookupTable_cuda[y5];
k = tid * 3;
j = tidd * nDstWidth / 2;
dpDst[j + k + 0] = lookupTable_cuda[u0];
dpDst[j + k + 1] = lookupTable_cuda[u1];
dpDst[j + k + 2] = lookupTable_cuda[u2];
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
dpDst[j + k + 0] = lookupTable_cuda[v0];
dpDst[j + k + 1] = lookupTable_cuda[v1];
dpDst[j + k + 2] = lookupTable_cuda[v2];
}
}
else if (scale == 6) {
uint32_t v0, y0, u0, y1;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 2;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 16;
pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

k = tid * 2;
j = tidd * nDstWidth;
dpDst[j + k + 0] = lookupTable_cuda[y0];
dpDst[j + k + 1] = lookupTable_cuda[y1];
k = tid;
j = tidd * nDstWidth / 2;
dpDst[j + k + 0] = lookupTable_cuda[u0];
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
dpDst[j + k + 1] = lookupTable_cuda[v0];
}
}
else if (scale == 2) {
uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
int nDstH = nDstHeight;
int nDstW = nDstWidth / 6;
if (tid < nDstW && tidd < nDstH) {
int j = tidd * nSrcPitch * scale;
int k = tid * 16;
pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
u0 = (uint32_t)(pF.x & 0x000003FF);
y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
y2 = (uint32_t)(pF.w & 0x000003FF);

pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
pF.z = (uint32_t)p_Src[j + k + 12];
pF.w = (uint32_t)p_Src[j + k + 14];

y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
v2 = (uint32_t)(pF.z & 0x000003FF);
y5 = (uint32_t)(pF.w & 0x000003FF);

k = tid * 6;
j = tidd * nDstWidth;
dpDst[j + k + 0] = lookupTable_cuda[y0];
dpDst[j + k + 1] = lookupTable_cuda[y1];
dpDst[j + k + 2] = lookupTable_cuda[y2];
dpDst[j + k + 3] = lookupTable_cuda[y3];
dpDst[j + k + 4] = lookupTable_cuda[y4];
dpDst[j + k + 5] = lookupTable_cuda[y5];
k = tid * 3;
j = tidd * nDstWidth / 2;
dpDst[j + k + 0] = lookupTable_cuda[u0];
dpDst[j + k + 1] = lookupTable_cuda[u1];
dpDst[j + k + 2] = lookupTable_cuda[u2];
j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
dpDst[j + k + 0] = lookupTable_cuda[v0];
dpDst[j + k + 1] = lookupTable_cuda[v1];
dpDst[j + k + 2] = lookupTable_cuda[v2];
}
}
}