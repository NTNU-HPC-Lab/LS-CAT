#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel(double *vals, double *ans, int N, double mu)
{
// Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N)
ans[idx] = vals[idx] + mu;
}