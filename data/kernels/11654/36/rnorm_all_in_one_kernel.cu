#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rnorm_all_in_one_kernel(float *vals, int n, float mu, float sigma)
{
// Usual block/thread indexing...
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int idx = myblock * blocksize + subthread;

// Setup the RNG:
hiprandState rng_state;
hiprand_init(9131 + idx*17, 0, 0, &rng_state);

if (idx < n) {
vals[idx] = mu + sigma * hiprand_normal(&rng_state);
}
return;
}