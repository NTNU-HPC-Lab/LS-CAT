#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double dnorm(float x, float mu, float sigma)
{
float std = (x - mu)/sigma;
float e = exp( - 0.5 * std * std);
return(e / ( sigma * sqrt(2 * 3.141592653589793)));
}
__global__ void truncNorm(float *out, float *unifVals, int N)
{
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;
if(idx < N) {
// 0.3413447460685 is pnorm(1) - pnorm(0), i.e. Pr( 0 <= Z <= 1) for Z ~ N(0, 1)
out[idx] = unifVals[idx] * dnorm(unifVals[idx], 0, 1)/0.3413447460685;
}
}