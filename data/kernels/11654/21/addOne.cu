#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOne(double* vals, int N)
{
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N) {
vals[idx] = vals[idx] + 1.0;
}
}