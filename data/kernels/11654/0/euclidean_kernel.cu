#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

// If an element is to be computed
if(x < n_a && y < n_b) {

__shared__ float temp[NUM_THREADS];

temp[threadIdx.x] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
temp[threadIdx.x] += (t * t);
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
d[y * pitch_d + x] = sqrt(temp[0]);
}
}
}