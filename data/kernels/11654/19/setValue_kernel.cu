#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setValue_kernel(int *vals, int N)
{
// Taken from
//geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
/* find my thread */
int idx = myblock*blocksize+subthread;

//	int idx = gridIdx.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
if(idx < N)
vals[idx] = idx;

}