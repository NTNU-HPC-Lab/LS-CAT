#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void euclideanDistance(const float *data_a, int nrow_a, const float *data_b, int nrow_b, int ncol, float *ans)
{
/*
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
*/

/* This is very simple, limited to 512 pairs of observations in total. */
int i = threadIdx.x, j = threadIdx.y;

if(i < nrow_a && j < nrow_b) {
int off_a = i,
off_b = j;
float sum = 0;
for(int k = 0; k < ncol ; k++, off_a += nrow_a, off_b+= nrow_b) {
float tmp = (data_a[off_a] - data_b[off_b]);
sum += tmp * tmp;
}
ans[i + j*nrow_a] = sqrt(sum);
}

}