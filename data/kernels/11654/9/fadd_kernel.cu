#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fadd_kernel(float *vals, int N, float mu)
{
// Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N)
vals[idx] = mu;
}