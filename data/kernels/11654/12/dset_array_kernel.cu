#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dset_array_kernel(double *vals, int N, double *mu)
{
// Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N)
vals[idx] = mu[0];
}