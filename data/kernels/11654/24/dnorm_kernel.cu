#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double dnorm(float x, float mu, float sigma)
{
float std = (x - mu)/sigma;
float e = exp( - 0.5 * std * std);
return(e / ( sigma * sqrt(2 * 3.141592653589793)));
}
__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < N) {
vals[idx] = sigma;
}
}