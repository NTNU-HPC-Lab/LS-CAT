#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOneFloat(double* vals, int N, float *out)
{
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N) {
out[idx] = (float) vals[idx] + 1.0;
}
}