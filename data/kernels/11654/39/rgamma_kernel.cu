#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgamma_kernel(hiprandState *state, int state_len, float *vals, int n, float a, float scale)
{
float sqrt32 = 5.656854;
float exp_m1 = 0.36787944117144232159;
float q1 = 0.04166669;
float q2 = 0.02083148;
float q3 = 0.00801191;
float q4 = 0.00144121;
float q5 = -7.388e-5;
float q6 = 2.4511e-4;
float q7 = 2.424e-4;
float a1 = 0.3333333;
float a2 = -0.250003;
float a3 = 0.2000062;
float a4 = -0.1662921;
float a5 = 0.1423657;
float a6 = -0.1367177;
float a7 = 0.1233795;

// Maybe want each thread to generate multiple...
// -- Would cut down on bloat from hiprandState *state,
// -- Would increase computation per thread + scalability
//
// state_len    :: length of rng states (one thread per rng_state)
//                 if (idx >= state_len){ do nothing... }
//
// n_per_thread :: number of rng's to generate per thread
//                 heurestically, each thread computes
//                 n_per_thread = n / state_len
//                 variates. However, if n_per_thread is
//                 not an integer, then some do one less,
//                 some one more.
//
// n(idx)       :: number of rng's that thread idx will generate
//
// n_lo         :: minimum number of rng's that a (used) thread
//                 will generate
//
// n_hi         :: maximum number of rng's that a (used) thread
//                 will generate
//
// n_extra      :: (= n % state_len) the number of threads that
//                 will generate n_hi rng's
//
// n_regular    :: (= state_len - n_extra) the number of threads that
//                 will generate n_lo rng's
//
// Note:
//
// n = (n_lo * n_regular) + (n_hi * n_extra)
//
// e.g.,
//
// n = 108, state_len = 40
// => n_lo = 2, n_hi = 3
// => n_extra = 28, n_regular = 12
// => threads  1-28 generate 3 rng's
// => threads 29-40 generate 2 rngs'
//
// thread idx writes to:
//
// if (idx < n_extra){
//   // 'extra' thread:
//   n_hi*idx : ((n_hi*idx) + (n_hi-1))
// }
// if (idx >= n_extra){
//   // 'regular' thread:
//   (n_hi*n_extra) + (n_lo*(idx-n_extra)):((n_lo*(idx-n_extra)) + (n_lo-1))
// }
//
// e.g., (cont...)
//
// thread  0 writes to: 0,1,2
// thread  1 writes to: 3,4,5
// ...
// thread 27 writes to: 81,82,83
// thread 28 writes to: 84,85
// thread 29 writes to: 86,87
// ...
// thread 40 writes to: 118,119
//

// Usual block/thread indexing...
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int idx = myblock * blocksize + subthread;

if (idx >= state_len) {
return;
}

// Number of regular and extra threads:
int n_extra = (n % state_len);
//int n_regular = n - n_extra;
int n_lo = (int) (n/state_len);
int n_hi = (n_lo+1);
int out_idx_start, n_per_thread;
if (idx < n_extra){
n_per_thread = n_hi;
out_idx_start = n_hi*idx;
} else {
n_per_thread = n_lo;
out_idx_start = n_hi*n_extra + n_lo*(idx-n_extra);
}
int gen_num;

// State variables:
float aa = 0.0;
float aaa = 0.0;
float s, s2, d;
float q0, b, si, c;
float e, p, q, r, t, u, v, w, x, ret_val;

// Generate RNG's for this thread...
for (gen_num=0; gen_num<n_per_thread; gen_num++){

// Note: Error checks on arguments removed!
if (a < 1.0) { // GS algorithm for parameters a < 1
if (a == 0){
vals[out_idx_start+gen_num] = 0.0;
continue;
}
e = 1.0 + exp_m1 * a;
while (1){
p = e * hiprand_normal(&state[idx]);
if (p >= 1.0) {
x = -logf((e - p)/a);
if (logf(hiprand_uniform(&state[idx])) <= (1.0 - a) * logf(x)){
break;
}
} else {
x = expf(logf(p) / a);
if (logf(hiprand_uniform(&state[idx])) <= x){
break;
}
}
}
vals[out_idx_start+gen_num] = scale * x;
continue;
}

// --- a >= 1 : GD algorithm ---

// Step 1: Recalculations of s2, s, d if a has changed
if (a != aa) {
aa = a;
s2 = a - 0.5;
s = sqrtf(s2);
d = sqrt32 - s * 12.0;
}

// Step 2: t = standard normal deviate,
//           x = (s,1/2) -normal deviate.

//printf("Step 2...\n");

// immediate acceptance (i)
t = hiprand_normal(&state[idx]);
x = s + 0.5 * t;
ret_val = x * x;
if (t >= 0.0){
vals[out_idx_start+gen_num] = scale * ret_val;
continue;
}

//printf("Step 3...\n");

// Step 3: u = 0,1 - uniform sample. squeeze acceptance (s)
u = hiprand_uniform(&state[idx]);
if ((d*u) <= (t*t*t)){
vals[out_idx_start+gen_num] = scale * ret_val;
continue;
}

//printf("Step 4...\n");

// Step 4: recalculations of q0, b, si, c if necessary
if (a != aaa){
aaa = a;
r = 1.0 / a;
q0 = ((((((q7 * r + q6) * r + q5) * r + q4) * r + q3) * r
+ q2) * r + q1) * r;

// Approximation depending on size of parameter a
// The constants in the expressions for b, si and c
// were established by numerical experiments

if (a <= 3.686) {
b = 0.463 + s + 0.178 * s2;
si = 1.235;
c = 0.195 / s - 0.079 + 0.16 * s;
} else if (a <= 13.022) {
b = 1.654 + 0.0076 * s2;
si = 1.68 / s + 0.275;
c = 0.062 / s + 0.024;
} else {
b = 1.77;
si = 0.75;
c = 0.1515 / s;
}

}

//printf("Step 5...\n");

// Step 5: no quotient test if x not positive
if (x > 0.0) {

// Step 6: calculation of v and quotient q
v = t / (s + s);
if (fabs(v) <= 0.25){
q = q0 + 0.5 * t * t * ((((((a7 * v + a6) * v + a5) * v + a4) * v
+ a3) * v + a2) * v + a1) * v;
} else {
q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
}

// Step 7: quotient acceptance (q)
if (logf(1.0 - u) <= q){
vals[out_idx_start+gen_num] = scale * ret_val;
continue;
}

}

//printf("Step 8...\n");

while (1){

// Step 8: e = standard exponential deviate
//	u =  0,1 -uniform deviate
//	t = (b,si)-double exponential (laplace) sample
e = -logf(hiprand_uniform(&state[idx])); // Expo(1)
u = hiprand_uniform(&state[idx]);
u = u + u - 1.0;
if (u < 0.0){
t = b - si * e;
} else {
t = b + si * e;
}
// Step	 9:  rejection if t < tau(1) = -0.71874483771719
if (t >= -0.71874483771719) {
// Step 10:	 calculation of v and quotient q
v = t / (s + s);
if (fabs(v) <= 0.25){
q = q0 + 0.5 * t * t *
((((((a7 * v + a6) * v + a5) * v + a4) * v + a3) * v
+ a2) * v + a1) * v;
} else {
q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
}
// Step 11:	 hat acceptance (h)
// (if q not positive go to step 8)
if (q > 0.0) {
w = expm1f(q); // expm1 is double prec...
//  ^^^^^ original code had approximation with rel.err < 2e-7
// if t is rejected sample again at step 8
if (c * fabs(u) <= w * expf(e - 0.5 * t * t)){
break;
}
}
}

} // repeat .. until  `t' is accepted

//printf("Returning at end...\n");

x = s + 0.5 * t;
vals[out_idx_start+gen_num] = scale * x * x;
continue;

} // end gen_num loop

return;
}