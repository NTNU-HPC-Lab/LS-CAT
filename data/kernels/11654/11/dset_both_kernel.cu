#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dset_both_kernel(double *vals, int N, double mu, float sd)
{
// Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
/* how big is each block within a grid */
int blocksize = blockDim.x * blockDim.y * blockDim.z;
/* get thread within a block */
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

int idx = myblock * blocksize + subthread;

if(idx < N)
vals[idx] = mu + sd;
}