#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void canberra_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
size_t x = blockIdx.x;
size_t y = blockIdx.y;

// If all element is to be computed
if(x < n_a && y < n_b) {
__shared__ float temp[NUM_THREADS];

temp[threadIdx.x] = 0.0;
for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
float num = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
float den = abs(vg_a[x * pitch_a + offset] + vg_b[y * pitch_b + offset]);
if(den != 0.0) {
temp[threadIdx.x] += num / den;
}
}

// Sync with other threads
__syncthreads();

// Reduce
for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
if(threadIdx.x < stride) {
temp[threadIdx.x] += temp[threadIdx.x + stride];
}
__syncthreads();
}
// Write to global memory
if(threadIdx.x == 0) {
d[y * pitch_d + x] = temp[0];
}
}
}