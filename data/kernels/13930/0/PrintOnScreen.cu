
#include <hip/hip_runtime.h>
void PrintTab()
{
	printf("\t");
}

void PrintNL()
{
	printf("\n");
}

void PrintTable(unsigned int** table, int rowsCount, int rowSize)
{
	int i,j;
	for(i=0; i < rowsCount; ++i)
	{
		for(j = 0; j < rowSize; ++j)
		{
			if(j % rowSize == 0 && i != 0)
				printf("\n");
			
				printf("%d ",table[i][j]);
		}
	}

	printf("\n\n");
}

void PrintSeparatorLine()
{
	printf("------------------------------------------------\n\n");
}
