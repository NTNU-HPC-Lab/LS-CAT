#include "hip/hip_runtime.h"
#include <math.h>
#include <limits.h>

extern unsigned int* gIntVector;
extern bool* gBoolPacket;
extern unsigned int** gIntTable;

unsigned int** MakeBitmap(int** inputSets, int* definiedItems); //implemented in MakeBitmap.cu file
bool** MakeBitmapBool(int** inputSets, int* definiedItems); //implemented in MakeBitmap.cu file

/*
int GetIntMaxBit()
{
	int intMax = INT_MAX;

	int bitCount = 0;
	while(intMax != 1)
	{
		intMax /= 2;
		++bitCount;
	}

	printf("max int: %d\n", INT_MAX);
	printf("bit: %d\n", ++bitCount);
	return 0;
}*/

unsigned int MakeIntFromBool(bool* boolPacket, int rowsize)
{
	if(rowsize > NUMBER_OF_BITS_IN_INT)
		exit(-1); //error

	unsigned int intValue = 0;

	int i;
	int boolValue = 0;
	for(i = 0; i < rowsize; ++i)
	{
		(boolPacket[i] == true) ? boolValue = 1 : boolValue = 0;
		
		intValue += pow(2.0f,i) * boolValue;
	}

	return intValue;
}

unsigned int* MakeIntVectorFromBitVector(bool* bitmapRow)
{
	//unsigned int* intVector = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));
	//bool* boolPacket = (bool*)malloc(NUMBER_OF_BITS_IN_INT * sizeof(bool));

	//boolPacket[0] = bitmapRow[0];

	int i, j;
	int boolPacketSize = NUMBER_OF_BITS_IN_INT;
	for(i = 0; i < NUMBER_OF_INTS_IN_ROW; ++i)
	{
		for(j = (i * NUMBER_OF_BITS_IN_INT); j < (i * NUMBER_OF_BITS_IN_INT + NUMBER_OF_BITS_IN_INT); ++j)
		{
			gBoolPacket[j - (i * NUMBER_OF_BITS_IN_INT)] = bitmapRow[j];
		}

		if( i == (NUMBER_OF_INTS_IN_ROW -1) )
		{
			boolPacketSize = NUMBER_OF_TRANSACTIONS % NUMBER_OF_BITS_IN_INT;

			//dla liczby transakcji <= 32
			if(boolPacketSize == 0 && NUMBER_OF_TRANSACTIONS > 0)
			{
				boolPacketSize = NUMBER_OF_BITS_IN_INT;
			}
		}

		gIntVector[i] = MakeIntFromBool(gBoolPacket, boolPacketSize);
	}

	return gIntVector;
}

unsigned int** MakeIntTableFromBitmap(bool** bitmap)
{
	int i;
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
	{
		*gIntTable[i] = *(MakeIntVectorFromBitVector(bitmap[i]));
	}

	return gIntTable;
}

//------------------ Test ------------------//

void TestMakeIntFromBool()
{
	PrintTestStartMessage("MakeIntFromBool");

	bool* boolRow = (bool*)malloc(32*sizeof(bool));
	int i;
	for(i = 0; i < 32; ++i)
		boolRow[i] = true;

	int intValue = MakeIntFromBool(boolRow, 32);
	printf("intValue: %u\n", intValue);

	PrintTestPassedMessage("MakeIntFromBool");
}

void TestMakeIntVectorFromBitVector()
{
	PrintTestStartMessage("MakeIntVectorFromBitVector");

	int** setsFromFile = GetSetsFromFile();
	int* definiedItems = GetDefinedItems();

	bool** bitmap = MakeBitmapBool(setsFromFile, definiedItems);

	unsigned int* intVector = (unsigned int*)malloc(sizeof(unsigned int*));
	unsigned int* intVector2 = (unsigned int*)malloc(sizeof(unsigned int*));

	*intVector = *(MakeIntVectorFromBitVector(bitmap[0]));
	*intVector2 = *(MakeIntVectorFromBitVector(bitmap[1]));

	unsigned int test1= intVector[0];
	unsigned int test2= intVector2[0];

	int intCount = NUMBER_OF_TRANSACTIONS / NUMBER_OF_BITS_IN_INT;
	if( (NUMBER_OF_TRANSACTIONS % NUMBER_OF_BITS_IN_INT) != 0 )
		++intCount;

	int i;
	for(i = 0; i < intCount; ++i)
	{
		PrintTab();

		printf("%u ", intVector[i]);

		PrintNL();
	}

	for(i = 0; i < intCount; ++i)
	{
		PrintTab();

		printf("%u ", intVector2[i]);

		PrintNL();
	}

	intVector[0] = intVector[0] & intVector2[0];

	for(i = 0; i < intCount; ++i)
	{
		PrintTab();

		printf("%u ", intVector[i]);

		PrintNL();
	}

	PrintTestPassedMessage("MakeIntVectorFromBitVector");
}

void TestMakeIntTableFromBitmap()
{
	PrintTestStartMessage("MakeIntTableFromBitmap");

	int** setsFromFile = GetSetsFromFile();
	int* definiedItems = GetDefinedItems();
	bool** bitmap = MakeBitmapBool(setsFromFile, definiedItems);

	unsigned int** intTable = MakeIntTableFromBitmap(bitmap);

	int i, j;

	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
	{
		PrintTab();
		for(j = 0; j < NUMBER_OF_INTS_IN_ROW; ++j)
		{
			printf("%d ", bitmap[i][j]);
		}
		PrintNL();
	}

	PrintNL();

	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
	{
		PrintTab();

		for(j = 0; j < NUMBER_OF_INTS_IN_ROW; ++j)
		{
			printf("%u ", intTable[i][j]);
		}

		PrintNL();
	}

	PrintTestPassedMessage("MakeIntTableFromBitmap");
}

//------------------ End of Test ------------------//
