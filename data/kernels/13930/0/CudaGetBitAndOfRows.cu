#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaGetBitAndOfRows(unsigned int* table1D, unsigned int* row, int rowSize, int tableRowCount)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < tableRowCount * rowSize)
{
table1D[idx] = table1D[idx] & row[idx % rowSize];
}
}