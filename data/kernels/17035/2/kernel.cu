#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *id, float *od, int w, int h, int depth)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int z = blockIdx.z * blockDim.z + threadIdx.z;
const int dataTotalSize   = w * h * depth;
const int radius		  = 2;
const int filter_size	  = 2*radius + 1;
const int sW			  = 6;				/* sW == 2 * filter_radius + blockDim.x (or same as 2 * filter_radius + blockDim.y) */
/* boarder do not concerned */
if(x >= w || y >= h || z >= depth)
return;
else
{
//global defined
int idx = z*w*h+y*w+x;

//3d grid(blocks) 2d block(threads)
int threadsPerBlock = blockDim.x * blockDim.y;
int blockId		    = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId	    = (blockId * threadsPerBlock)
+ (threadIdx.y * blockDim.x) + threadIdx.x;
int g_Idx			= threadId;

//2d shared memory working
__shared__ unsigned char smem[sW][sW];
int s_Idx = threadIdx.x + (threadIdx.y * sW);
int s_IdxY = s_Idx / sW;
int s_IdxX = s_Idx % sW;

//Here: definition error, need edit, haven't finished yet.
//int g_IdxY = s_IdxY + (blockIdx.y * blockDim.y);
//int g_IdxX = s_IdxX + (blockIdx.x * blockDim.x);
//int g_Idx  = g_IdxX + (g_IdxY * w);

//32 threads working together per warp
if(s_IdxY < sW && s_IdxX < sW)	//Here: boarder concerned error, need edit
{
if(x >= 0 && y < w && y >= 0 && y < h && z >= 0 && z < depth )	//Here: boarder concerned error, need edit
smem[s_IdxY][s_IdxX] = id[g_Idx];
else
smem[s_IdxY][s_IdxX] = 0;
__syncthreads();
}

/*compute the sum using shared memory*/
float avg = 0.0;
for (int i = -radius; i <= radius; i++){
if(s_IdxY + i < 0 /*|| g_IdxY > h*/ )			//Here: boarder concerned error, need edit
avg += 0.0;
else
avg += smem[s_IdxY+i][s_IdxX];
}

/*register to global, by now thread*/
avg /= filter_size;
if(idx < dataTotalSize)
od[idx] = avg;
}
}