#include "hip/hip_runtime.h"
#include "includes.h"
/*
Vector addition with a single thread for each addition
*/



/*
Vector addition with thread mapping and thread accessing its neighbor parallely
*/

//slower than simpler


/*
Matrix Matrix multiplication with a single thread for each row
*/


/*
Matrix Matrix multiplication with a single thread for each result element
*/


/*
Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/


__global__ void good_addition(int *a, int *b, int *c, int len)
{
int tid= threadIdx.x + blockIdx.x * blockDim.x;
const int thread_count= blockDim.x*gridDim.x;
int step = len/thread_count;

int start_index = tid*step;
int end_index= (tid+1)* step;
if (tid==thread_count-1) end_index=len;
//printf("Step is %d\n",step);
while(start_index< end_index)
{
c[start_index]=a[start_index]+b[start_index];

//printf("I am block: %d with tid: %d Result %d \n",blockIdx.x,tid,c[tid]);
start_index +=1;
}
}