#include "hip/hip_runtime.h"
#include "includes.h"
/*
Vector addition with a single thread for each addition
*/



/*
Vector addition with thread mapping and thread accessing its neighbor parallely
*/

//slower than simpler


/*
Matrix Matrix multiplication with a single thread for each row
*/


/*
Matrix Matrix multiplication with a single thread for each result element
*/


/*
Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/


__global__ void matrix_matrix_new(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
int tid= threadIdx.x + blockIdx.x *  blockDim.x;
int temp=0;
while(tid<n_row*n_col)
{
// find the row index of A
int i=tid / n_col;
// find the column index of B
int j=tid % n_col;
// multiply the row and column
temp=0;
for(int k=0;k<n_comm;k++)
{
temp+= a[i*n_comm+k]*b[j+k*n_col];
}
c[tid]=temp;
tid+= blockDim.x * gridDim.x;
}
}