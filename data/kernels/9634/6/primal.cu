#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void primal(float *y1, float *y2, float *xbar, float sigma, int w, int h, int nc) {
int x = threadIdx.x + blockDim.x * blockIdx.x;
int y = threadIdx.y + blockDim.y * blockIdx.y;

if (x < w && y < h) {
int i;
float x1, x2, val, norm;

for (int z = 0; z < nc; z++) {
i = x + w * y + w * h * z;

val = xbar[i];
x1 = (x+1<w) ? (xbar[(x+1) + w * y + w * h * z] - val) : 0.f;
x2 = (y+1<h) ? (xbar[x + w * (y+1) + w * h * z] - val) : 0.f;

x1 = y1[i] + sigma * x1;
x2 = y2[i] + sigma * x2;

norm = sqrtf(x1*x1+x2*x2);

y1[i] = x1 / fmax(1.f, norm);
y2[i] = x2 / fmax(1.f, norm);
}
}
}