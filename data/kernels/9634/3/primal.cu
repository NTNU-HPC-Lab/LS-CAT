#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void primal(float* u, float* u_, const float* f, const float* p1, const float* p2, const double tau, const int X, const int Y)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

// center point
int c = y*X + x;

float div_x = 0.0f;
float div_y = 0.0f;

if (x == 0)
div_x = p1[c];
if (x > 0 & x < X-1)
div_x = p1[c]-p1[c-1];
if (x == X-1)
div_x = -p1[c-1];

if (y == 0)
div_y = p2[c];
if (y > 0 && y < Y-1)
div_y = p2[c]-p2[c-X];
if (y == Y-1)
div_y = -p2[c-X];

float u_old = u[c];
u[c] = (u_old + tau*(+div_x+div_y+f[c]))/(1+tau);
u_[c] = 2*u[c]-u_old;
}