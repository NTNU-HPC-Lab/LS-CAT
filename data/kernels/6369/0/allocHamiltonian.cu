#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CudaOperations.cu
*
*  Created on: Feb 6, 2019
*      Author: alexander
*/


__global__ void allocHamiltonian(float* devMat, float* devSpins, int index, int size, double* energyTempor) {
int i;
int j;

int wIndex = threadIdx.x + blockIdx.x * blockDim.x;
while (wIndex < size * size) {
i = wIndex % size;
j = (int) (wIndex / size);
energyTempor[wIndex] = (double) (devSpins[i + index * size]
* devSpins[j + index * size] * devMat[wIndex]);
wIndex = wIndex + blockDim.x * gridDim.x;
}
}