#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CudaOperations.cu
*
*  Created on: Feb 6, 2019
*      Author: alexander
*/


__global__ void quickSum(double* energyTempor, int size) {
long long offset = 1;
int wIndex;
while (offset < size * size) {
wIndex = threadIdx.x;
while ((wIndex * 2 + 1) * offset < size * size) {
energyTempor[wIndex * 2 * offset] += energyTempor[(wIndex * 2 + 1)
* offset];
wIndex = wIndex + blockDim.x;
}
offset *= 2;
__syncthreads();
}
}