#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CudaOperations.cu
*
*  Created on: Feb 6, 2019
*      Author: alexander
*/


__global__ void cudaKernelPull(float* mat, float* spins, int size, float* temp, float tempStep, float* meanFieldElements, bool* continueIteration, float minDiff, int* unemptyCells, float linearCoef) {
int blockId = blockIdx.x;
int thrId = threadIdx.x;

do {
// Lessen temperature
if (thrId == 0)
temp[blockId] = temp[blockId] - tempStep;

// Stabilize
do {
__syncthreads();
// By default current iteration is the last one
if (thrId == 0)
continueIteration[blockId] = false;

for (int spinId = 0; spinId < size; ++spinId) {
__syncthreads();

// Transitional value assignment
int wIndex = thrId;
while (wIndex < unemptyCells[spinId * (size + 1)]) {
meanFieldElements[wIndex + blockId * size] =
spins[unemptyCells[spinId * (size + 1) + wIndex + 1]
+ blockId * size]
* mat[spinId * size
+ unemptyCells[spinId * (size + 1)
+ wIndex + 1]];
// BEWARE: Matrix is symmetrical!
wIndex = wIndex + blockDim.x;
}
__syncthreads();

// Parallelized mean-field computation
long long offset = 1;
while (offset < unemptyCells[spinId * (size + 1)]) {
wIndex = thrId;
while ((wIndex * 2 + 1) * offset
< unemptyCells[spinId * (size + 1)]) {
meanFieldElements[wIndex * 2 * offset + blockId * size] +=
meanFieldElements[(wIndex * 2 + 1) * offset
+ blockId * size];
wIndex = wIndex + blockDim.x;
}
offset *= 2;
__syncthreads();
}
__syncthreads();

// Mean-field calculation complete - write new spin and delta
if (thrId == 0) {
float meanField = meanFieldElements[blockId * size];
float old = spins[spinId + blockId * size];
if (temp[blockId] > 0) {
spins[spinId + blockId * size] = -1
* tanh(meanField / temp[blockId]) * linearCoef
+ spins[spinId + blockId * size]
* (1 - linearCoef);
} else if (meanField > 0)
spins[spinId + blockId * size] = -1;
else
spins[spinId + blockId * size] = 1;

if (minDiff < fabs(old - spins[spinId + blockId * size]))
continueIteration[blockId] = true; // Too big delta. One more iteration needed
}
__syncthreads();
}
} while (continueIteration[blockId]);
} while (temp[blockId] >= 0);
}