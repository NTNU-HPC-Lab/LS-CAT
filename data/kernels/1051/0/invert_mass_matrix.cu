#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void invert_mass_matrix(double *values, unsigned int size)
{
unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i < size)
{
if (values[i] > 1e-15)
values[i] = 1. / values[i];
else
values[i] = 0.;
}
}