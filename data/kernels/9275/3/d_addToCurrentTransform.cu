#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_addToCurrentTransform(float* d_currentTransform, float* d_invViewMatrix) {
float result[12] = {0.f};
for (int i = 0; i < 3; ++i) {
for (int j = 0; j < 4; ++j) {
for (int k = 0; k < 4; ++k) {
result[i * 4 + j] += d_invViewMatrix[i * 4 + k] * d_currentTransform[k * 4 + j];
}
}
}
for (int i = 0; i < 12; ++i) {	// The last row of currentTransform remains (0,0,0,1)
d_currentTransform[i] = result[i];
}
}