#include "hip/hip_runtime.h"
#include "includes.h"



#define BLOCK_SIZE 16

__global__ void MultiplyGPU(float* a, float* b, float* c,int t)
{

int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

float aux =0;

if (i < t)
{
if (j < t)
{
for (int k = 0; k < t; k++)
{
aux += a[i * t + k] * b[k * t + j];
}

c[i * t + j] = aux;
}

}

}