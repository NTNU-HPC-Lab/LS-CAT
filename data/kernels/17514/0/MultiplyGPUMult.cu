#include "hip/hip_runtime.h"
#include "includes.h"



#define BLOCK_SIZE 16

__device__ float* GetSubMatrix(float * a, int tam, int row, int col)
{
float* aSub;
aSub = &a[tam * BLOCK_SIZE * row + BLOCK_SIZE * col];
return aSub;
}
__global__ void MultiplyGPUMult(float * a, float *b, float *c,int t)
{
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

float* Csub = GetSubMatrix(c, t, blockRow, blockCol);

float Cvalue = 0;

int row = threadIdx.y;
int col = threadIdx.x;

for (int m = 0; m < t / BLOCK_SIZE; m++)
{
float* Asub = GetSubMatrix(a, t, blockRow, m);
float* Bsub = GetSubMatrix(b, t, m, blockCol);

__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

As[row][col] = Asub[row * t + col];
Bs[row][col] = Bsub[row * t + col];

__syncthreads();

for (int e = 0; e < BLOCK_SIZE; e++)
{
Cvalue += As[row][e] * Bs[e][col];
}

__syncthreads();

}

Csub[row * t + col] = Cvalue;
}