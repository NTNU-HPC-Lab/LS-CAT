#include "hip/hip_runtime.h"
#include "includes.h"
//**********************************
//Nathan Durst
//FFT Cuda Program
//December, 5 2016
//**********************************
//This application uses cuda c and implements
// the Cooley-Tukey FFT algorithm to transforms
// an array of complex numbers into a data set
// correlation of complex numbers.
#define N 16384
#define PI 3.14

//kernel function declaration

__global__ void FFT(float * R, float * I, float * xR, float * xI)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
float real = 0, imag = 0;

//iterate through entire array for each index and calculate even
// and odd for real and imaginary numbers.
for (int i = 0; i<(N/2); i++)
{
//even
real += R[i] * cos((2*PI*(i*2))/N) - I[i] * sin((2*PI*id*(i*2))/N);
imag += R[i] * -sin((2*PI*(i*2))/N) + I[i] * cos((2*PI*id*(i*2))/N);

//odd
real += R[i] * cos((2*PI*(i*2+1))/N) - I[i] * sin((2*PI*id*(i*2+1))/N);
imag += R[i] * -sin((2*PI*(i*2+1))/N) + I[i] * cos((2*PI*id*(i*2+1))/N);
}
xR[id] = real;
xI[id] = imag;
}