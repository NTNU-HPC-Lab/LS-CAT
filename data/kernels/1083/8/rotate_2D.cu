#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_2D(float* coords, size_t dim_y, size_t dim_x, float cos_angle, float sin_angle){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y;
float new_y, new_x;
float old_y = coords[index];
float old_x = coords[index + total];
if(index < total){
new_y = cos_angle * old_y + sin_angle * old_x;
new_x = -sin_angle * old_y + cos_angle * old_x;
__syncthreads();
coords[index] = new_y;
coords[index + total] = new_x;
__syncthreads();
}
}