#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_coords_2D(float* coords, size_t y, size_t x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t id_x = index % x;
size_t id_y = index / x;
if(index < x * y){
coords[id_x + id_y * x] = id_y - (float)y/2.0;
coords[id_x + id_y * x + x*y] = id_x - (float)x/2.0;
}
__syncthreads();
}