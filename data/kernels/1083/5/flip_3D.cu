#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void exchange(float &a, float &b){
float temp = a;
a = b;
b = temp;
}
__global__ void flip_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, int do_z, int do_y, int do_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y * dim_z;
size_t total_xy = dim_x * dim_y;
size_t id_x = index % dim_x;
size_t id_y = (index / dim_x) % dim_x;
size_t id_z = index / (dim_x * dim_y);
if(index < total){
if(do_x && id_x < (dim_x / 2)){
exchange(coords[2 * total + id_z * total_xy + id_y * dim_x + id_x],
coords[2 * total + id_z * total_xy + id_y * dim_x + dim_x-1 - id_x]);
__syncthreads();
}
if(do_y && id_y < (dim_y / 2)){
exchange(coords[total + id_z * total_xy + id_y * dim_x + id_x],
coords[total + id_z * total_xy + (dim_y-1 - id_y) * dim_x + id_x]);
__syncthreads();
}
if(do_z && id_z < (dim_z / 2)){
exchange(coords[id_z * total_xy + id_y * dim_x + id_x],
coords[(dim_z-1 -id_z) * total_xy + id_y * dim_x + id_x]);
__syncthreads();
}
}
}