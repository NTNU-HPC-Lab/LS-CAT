#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_random(float *random, size_t total_size){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < total_size){
random[index] = random[index] * 2.0 - 1.0;
__syncthreads();
}
}