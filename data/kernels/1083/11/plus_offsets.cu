#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void plus_offsets(float *coords, float *random, size_t total_size, float alpha){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < total_size){
coords[index] += random[index] * alpha;
__syncthreads();
}
}