#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void exchange(float &a, float &b){
float temp = a;
a = b;
b = temp;
}
__global__ void flip_2D(float* coords, size_t dim_y, size_t dim_x, int do_y, int do_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y;
size_t id_x = index % dim_x;
size_t id_y = index / dim_x;
if(index < total){
if(do_x && id_x < (dim_x / 2)){
exchange(coords[total + id_y * dim_x + id_x],
coords[total + id_y * dim_x + dim_x-1 - id_x]);
__syncthreads();
}
if(do_y && id_y < (dim_y / 2)){
exchange(coords[id_y * dim_x + id_x], coords[(dim_y-1 - id_y) * dim_x + id_x]);
__syncthreads();
}
}
}