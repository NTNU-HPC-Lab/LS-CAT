#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void device_only_copy(float* output, float* input, size_t total_size){
for(size_t i = blockIdx.x * blockDim.x + threadIdx.x;
i < total_size;
i += blockDim.x * gridDim.x){
output[i] = input[i];
}
__syncthreads();
}