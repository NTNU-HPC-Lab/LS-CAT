#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, float* rot_matrix){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y * dim_z;
float new_y = 0, new_x = 0, new_z = 0;
float old_z = coords[index];
float old_y = coords[index + total];
float old_x = coords[index + 2 * total];
if(index < total){
new_z = old_z * rot_matrix[0] + old_y * rot_matrix[3] + old_x * rot_matrix[6];
new_y = old_z * rot_matrix[1] + old_y * rot_matrix[4] + old_x * rot_matrix[7];
new_x = old_z * rot_matrix[2] + old_y * rot_matrix[5] + old_x * rot_matrix[8];
__syncthreads();
coords[index] = new_z;
coords[index + total] = new_y;
coords[index + 2 * total] = new_x;
__syncthreads();
}
}