#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void translate_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, float seg_z, float seg_y, float seg_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y * dim_z;
if(index < total){
coords[index] += seg_z;
coords[index + total] += seg_y;
coords[index + total * 2] += seg_x;
__syncthreads();
}
}