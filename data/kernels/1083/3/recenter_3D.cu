#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void recenter_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y * dim_z;
if(index < total){
coords[index] += (float)dim_z/2.0;
coords[index + total] += (float)dim_y/2.0;
coords[index + 2 * total] += (float)dim_x/2.0;
}
__syncthreads();
}