#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void translate_2D(float* coords, size_t dim_y, size_t dim_x, float seg_y, float seg_x){
size_t index = blockIdx.x * blockDim.x + threadIdx.x;
size_t total = dim_x * dim_y;
if(index < total){
coords[index] += seg_y;
coords[index + total] += seg_x;
__syncthreads();
}
}