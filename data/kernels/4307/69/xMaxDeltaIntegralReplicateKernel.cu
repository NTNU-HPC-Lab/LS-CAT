#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralReplicateKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const int strideH, const int strideW) {

// TODO: use block dim instead
const int hOut = (h + strideH - 1) / strideH;
const int wOut = (w + strideW - 1) / strideW;

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int yOut = id % wOut; id /= wOut; // 0-indexed
const int xOut = id % hOut; id /= hOut; // 0-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

const int x = xOut*strideH + 1;
const int y = yOut*strideW + 1;

tmpArray += windowIdx * hOut * wOut;

// const int xMinInt = (int)ceil(xMin[windowIdx]-1);
const int yMinInt = (int)ceil(yMin[windowIdx]-1);
const int xMaxInt = (int)floor(xMax[windowIdx]);
const int yMaxInt = (int)floor(yMax[windowIdx]);

float delta = 0;

delta +=
intData[max(1,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(1,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMinInt, w))];
delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt, w))];

delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
tmpArray[xOut*wOut + yOut] = delta;
}
}