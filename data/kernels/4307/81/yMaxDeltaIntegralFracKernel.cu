#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMaxDeltaIntegralFracKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

int id = NUM_THREADS * blockIdx.x + threadIdx.x;
tmpArray += id; // tmpArray now points to our output pixel

const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int windowIdx = id % nWindows; id /= nWindows;

// `id` is now is now the current global input plane number
intData  += id * intDataStrideChannel;
inData   += id *  inDataStrideChannel;

const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
const int & batchIdx = id;

if (batchIdx < batchSize) {

const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);
const float xMinFrac = xMinInt-xMin[globalWindowIdx]+1;

// const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
// const float yMinFrac = yMinInt-yMin[globalWindowIdx]+1;

const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
const float xMaxFrac = xMax[globalWindowIdx]-xMaxInt;

const int yMaxInt = (int)floor(yMax[globalWindowIdx]);
// const float yMaxFrac = yMax[globalWindowIdx]-yMaxInt;

int valid;

valid = not (y+yMaxInt >= w) & not (x+xMinInt < 1) & not (x+xMinInt > h);
const float trCorner = valid * inData[
max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
max(0,min(w-1,y+yMaxInt  ))];

valid = not (y+yMaxInt >= w) & not (x+xMaxInt < 0) & not (x+xMaxInt >= h);
const float brCorner = valid * inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMaxInt  ))];

float delta = 0;

delta += trCorner * xMinFrac;
delta += brCorner * xMaxFrac;

delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt+1, w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];
delta -=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMaxInt+1, w))];
delta +=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];

delta *= (y+yMaxInt >= 0) & (y+yMaxInt < w);
*tmpArray = delta;
}
}