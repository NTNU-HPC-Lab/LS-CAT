#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dirtyFixWindowsVarScaleKernel( float *xMin, float *xMax, float *yMin, float *yMax, const int size, const float h, const float w, const float minWidth) {

int idx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

if (idx < 2*size) {
float paramMin, paramMax;

if (idx < size) {
paramMin = max(-h+1, min(h-1, xMin[idx]));
paramMax = max(-h+1, min(h-1, xMax[idx]));

if (paramMin + minWidth - 0.99 > paramMax) {
const float mean = 0.5 * (paramMin + paramMax);
paramMin = mean - 0.5 * (minWidth - 0.9);
paramMax = mean + 0.5 * (minWidth - 0.9);
}

xMin[idx] = paramMin;
xMax[idx] = paramMax;
} else {
idx -= size;
paramMin = max(-w+1, min(w-1, yMin[idx]));
paramMax = max(-w+1, min(w-1, yMax[idx]));

if (paramMin + minWidth - 0.99 > paramMax) {
const float mean = 0.5 * (paramMin + paramMax);
paramMin = mean - 0.5 * (minWidth - 0.9);
paramMax = mean + 0.5 * (minWidth - 0.9);
}

yMin[idx] = paramMin;
yMax[idx] = paramMax;
}
}
}