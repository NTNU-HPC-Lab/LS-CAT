#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMinDeltaIntegralKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax) {

int id = NUM_THREADS * blockIdx.x + threadIdx.x;
tmpArray += id; // tmpArray now points to our output pixel

const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int windowIdx = id % nWindows; id /= nWindows;

// `id` is now is now the current global input plane number
intData  += id * intDataStrideChannel;

const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
const int & batchIdx = id;

if (batchIdx < batchSize) {

const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);
const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
// const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
const int yMaxInt = (int)floor(yMax[globalWindowIdx]);

float delta = 0;

delta +=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMinInt-1, h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMinInt, w))];
delta +=
intData[max(0,min(x+xMinInt-1, h))*(w+1)
+ max(0,min(y+yMinInt, w))];

delta *= (x+xMinInt >= 1 and x+xMinInt < h);
*tmpArray = -delta;
}
}