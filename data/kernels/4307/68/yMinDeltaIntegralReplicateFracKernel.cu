#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMinDeltaIntegralReplicateFracKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const float *inData, const int inDataStrideRow, const int strideH, const int strideW) {

// TODO: use block dim instead
const int hOut = (h + strideH - 1) / strideH;
const int wOut = (w + strideW - 1) / strideW;

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int yOut = id % wOut; id /= wOut; // 0-indexed
const int xOut = id % hOut; id /= hOut; // 0-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

const int x = xOut*strideH + 1;
const int y = yOut*strideW + 1;

tmpArray += windowIdx * hOut * wOut;

const int xMinInt = (int)ceil(xMin[windowIdx]-1);
const float xMinFrac = xMinInt-xMin[windowIdx]+1;

const int yMinInt = (int)ceil(yMin[windowIdx]-1);
// const float yMinFrac = yMinInt-yMin[windowIdx]+1;

const int xMaxInt = (int)floor(xMax[windowIdx]);
const float xMaxFrac = xMax[windowIdx]-xMaxInt;

// const int yMaxInt = (int)floor(yMax[windowIdx]);
// const float yMaxFrac = yMax[windowIdx]-yMaxInt;

const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
inData[
max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];
const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];
// const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
//                      inData[
//                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
//                         max(0,min(w-1,y+yMaxInt  ))];
// const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
//                     inData[
//                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
//                         max(0,min(w-1,y+yMaxInt  ))];

float delta = 0;

delta += tlCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
delta += blCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

delta +=
intData[max(0,min(x+xMaxInt, h))*(w+1)
+ max(0,min(y+yMinInt  , w))];
delta -=
intData[max(0,min(x+xMaxInt, h))*(w+1)
+ max(0,min(y+yMinInt-1, w))];
delta -=
intData[max(0,min(x+xMinInt, h))*(w+1)
+ max(0,min(y+yMinInt  , w))];
delta +=
intData[max(0,min(x+xMinInt, h))*(w+1)
+ max(0,min(y+yMinInt-1, w))];

delta *= (y+yMinInt >= 1 and y+yMinInt < w);
tmpArray[xOut*wOut + yOut] *= -delta;
}
}