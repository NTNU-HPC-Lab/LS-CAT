#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMinDeltaIntegralFracKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow) {

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and x <= h and y <= w) {

tmpArray += windowIdx * h * w;

const int rem = windowIdx % 4;

if (rem == 0) {
tmpArray[(x-1)*w + (y-1)] = 0;
} else {

const float xMinStretched = rem == 0 ? -h :
xMin[3*(windowIdx/4) + (rem > 0 ? (rem-1) : rem)];
// const float xMaxStretched = rem == 1 ?  h :
//     xMax[3*(windowIdx/4) + (rem > 1 ? (rem-1) : rem)];
const float yMinStretched = rem == 2 ? -w :
yMin[3*(windowIdx/4) + (rem > 2 ? (rem-1) : rem)];
const float yMaxStretched = rem == 3 ?  w :
yMax[3*(windowIdx/4) + (rem > 3 ? (rem-1) : rem)];

const int xMinInt = (int)ceil(xMinStretched-1);
// const float xMinFrac = xMinInt-xMinStretched+1;

const int yMinInt = (int)ceil(yMinStretched-1);
const float yMinFrac = yMinInt-yMinStretched+1;

// const int xMaxInt = (int)floor(xMaxStretched);
// const float xMaxFrac = xMaxStretched-xMaxInt;

const int yMaxInt = (int)floor(yMaxStretched);
const float yMaxFrac = yMaxStretched-yMaxInt;

const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
inData[
max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];
// const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
//                     inData[
//                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
//                         max(0,min(w-1,y+yMinInt-1))];
const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
inData[
max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
max(0,min(w-1,y+yMaxInt  ))];
// const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
//                     inData[
//                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
//                         max(0,min(w-1,y+yMaxInt  ))];

float delta = 0;

delta += trCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
delta += tlCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

delta +=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMinInt-1, h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMinInt, w))];
delta +=
intData[max(0,min(x+xMinInt-1, h))*(w+1)
+ max(0,min(y+yMinInt, w))];

delta *= (x+xMinInt >= 1 and x+xMinInt < h);
tmpArray[(x-1)*w + (y-1)] *= -delta;
}
}
}