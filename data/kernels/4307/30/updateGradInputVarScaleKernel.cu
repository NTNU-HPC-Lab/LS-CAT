#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradInputVarScaleKernel( float *gradOutputIntData, float *gradInputData, int h, int w, int nWindows, float *xMin, float *xMax, float *yMin, float *yMax) {

const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

if (x < h and y < w) {

int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;
double outValue = 0;

for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

xMinCurr = (int)ceil(-xMax[windowIdx]);
yMinCurr = (int)ceil(-yMax[windowIdx]);

xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;

// The following code block implements these lines
// as if they were executed simultaneously (see `void updateGradInputFrac()`):
// xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
// xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
// yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
// yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

bool needToChangeMin, needToChangeMax;

needToChangeMin = x == 0   and xMaxCurr >= 0;
needToChangeMax = x == h-1 and xMinCurr <= 0;
if (needToChangeMin) xMinCurr = 0;
if (needToChangeMax) xMaxCurr = h+66;

needToChangeMin = y == 0   and yMaxCurr >= 0;
needToChangeMax = y == w-1 and yMinCurr <= 0;
if (needToChangeMin) yMinCurr = 0;
if (needToChangeMax) yMaxCurr = w+66;

const int t = max(0, min(x+xMinCurr, h) );
const int b = max(0, min(x+xMaxCurr, h) );
const int l = max(0, min(y+yMinCurr, w) );
const int r = max(0, min(y+yMaxCurr, w) );

outValue += gradOutputIntData[b*(w+1) + r];
outValue -= gradOutputIntData[t*(w+1) + r];
outValue -= gradOutputIntData[b*(w+1) + l];
outValue += gradOutputIntData[t*(w+1) + l];

// go to the next channel
gradOutputIntData += (h+1)*(w+1);
}

gradInputData[x*w + y] = outValue;
}
}