#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralFracKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

int id = NUM_THREADS * blockIdx.x + threadIdx.x;
tmpArray += id; // tmpArray now points to our output pixel

const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int windowIdx = id % nWindows; id /= nWindows;

// `id` is now is now the current global input plane number
intData  += id * intDataStrideChannel;
inData   += id *  inDataStrideChannel;

const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
const int & batchIdx = id;

if (batchIdx < batchSize) {

// const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);
// const float xMinFrac = xMinInt-xMin[globalWindowIdx]+1;

const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
const float yMinFrac = yMinInt-yMin[globalWindowIdx]+1;

const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
// const float xMaxFrac = xMax[globalWindowIdx]-xMaxInt;

const int yMaxInt = (int)floor(yMax[globalWindowIdx]);
const float yMaxFrac = yMax[globalWindowIdx]-yMaxInt;

int valid;

valid = not (y+yMinInt < 1) & not (y+yMinInt > w) & not (x+xMaxInt >= h);
const float blCorner = valid * inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];

valid = not (y+yMaxInt < 0) & not (y+yMaxInt >= w) & not (x+xMaxInt >= h);
const float brCorner = valid * inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMaxInt  ))];

float delta = 0;

delta += brCorner * yMaxFrac;
delta += blCorner * yMinFrac;

delta +=
intData[max(0,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];
delta -=
intData[max(0,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMinInt  , w))];
delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt  , w))];

delta *= (x+xMaxInt >= 0) & (x+xMaxInt < h);
*tmpArray = delta;
}
}