#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulateColsKernel(float *input, float *output, int channels, int h, int w) {
// global column index (of all `channels * w` columns in this image)
int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

if (colIdx < channels * w) {
// jump to current channel
input  += (colIdx / w) * h * w;
output += (colIdx / w) * (h+1) * (w+1);
colIdx %= w; // switch to local column index,
++colIdx;    // it's 1-indexed because first output column is always zero

output[colIdx] = 0; // first element of every column is always zero
double sum = 0;

for (int i = 1; i <= h; ++i) {
sum += static_cast<double>(input[(i-1) * w + colIdx - 1]);
output[i * (w+1) + colIdx] = static_cast<float>(sum);
}
}
}