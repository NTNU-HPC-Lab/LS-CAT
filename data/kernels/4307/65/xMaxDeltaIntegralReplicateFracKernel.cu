#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralReplicateFracKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int strideH, const int strideW) {

// TODO: use block dim instead
const int hOut = (h + strideH - 1) / strideH;
const int wOut = (w + strideW - 1) / strideW;

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int yOut = id % wOut; id /= wOut; // 0-indexed
const int xOut = id % hOut; id /= hOut; // 0-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

const int x = xOut*strideH + 1;
const int y = yOut*strideW + 1;

tmpArray += windowIdx * hOut * wOut;

// const int xMinInt = (int)ceil(xMin[windowIdx]-1);
// const float xMinFrac = xMinInt-xMin[windowIdx]+1;

const int yMinInt = (int)ceil(yMin[windowIdx]-1);
const float yMinFrac = yMinInt-yMin[windowIdx]+1;

const int xMaxInt = (int)floor(xMax[windowIdx]);
// const float xMaxFrac = xMax[windowIdx]-xMaxInt;

const int yMaxInt = (int)floor(yMax[windowIdx]);
const float yMaxFrac = yMax[windowIdx]-yMaxInt;

// const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
//                      inData[
//                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
//                         max(0,min(w-1,y+yMinInt-1))];
const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];
// const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
//                      inData[
//                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
//                         max(0,min(w-1,y+yMaxInt  ))];
const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMaxInt  ))];

float delta = 0;

delta += brCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
delta += blCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

delta +=
intData[max(0,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMinInt, w))];
delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt, w))];

delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
tmpArray[xOut*wOut + yOut] = delta;
}
}