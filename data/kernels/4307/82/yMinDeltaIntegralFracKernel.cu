#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMinDeltaIntegralFracKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

int id = NUM_THREADS * blockIdx.x + threadIdx.x;
tmpArray += id; // tmpArray now points to our output pixel

const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int windowIdx = id % nWindows; id /= nWindows;

// `id` is now is now the current global input plane number
intData  += id * intDataStrideChannel;
inData   += id *  inDataStrideChannel;

const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
const int & batchIdx = id;

if (batchIdx < batchSize) {

const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);
const float xMinFrac = xMinInt-xMin[globalWindowIdx]+1;

const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
// const float yMinFrac = yMinInt-yMin[globalWindowIdx]+1;

const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
const float xMaxFrac = xMax[globalWindowIdx]-xMaxInt;

// const int yMaxInt = (int)floor(yMax[globalWindowIdx]);
// const float yMaxFrac = yMax[globalWindowIdx]-yMaxInt;

int valid;

valid = not (y+yMinInt < 1) & not (x+xMinInt < 1) & not (x+xMinInt > h);
const float tlCorner = valid * inData[
max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];

valid = not (y+yMinInt < 1) & not (x+xMaxInt < 0) & not (x+xMaxInt >= h);
const float blCorner = valid * inData[
max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
max(0,min(w-1,y+yMinInt-1))];

float delta = 0;

delta += tlCorner * xMinFrac;
delta += blCorner * xMaxFrac;

delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt  , w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt-1, w))];
delta -=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMinInt  , w))];
delta +=
intData[max(0,min(x+xMinInt  , h))*(w+1)
+ max(0,min(y+yMinInt-1, w))];

delta *= (y+yMinInt >= 1) & (y+yMinInt <= w);
*tmpArray = -delta;
}
}