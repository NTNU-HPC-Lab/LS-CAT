#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMaxDeltaIntegralKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax) {

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and x <= h and y <= w) {

tmpArray += windowIdx * h * w;

const int xMinInt = (int)ceil(xMin[windowIdx]-1);
// const int yMinInt = (int)ceil(yMin[windowIdx]-1);
const int xMaxInt = (int)floor(xMax[windowIdx]);
const int yMaxInt = (int)floor(yMax[windowIdx]);

float delta = 0;

delta +=
intData[max(0,min(x+xMaxInt, h))*(w+1)
+ max(1,min(y+yMaxInt+1, w))];
delta -=
intData[max(0,min(x+xMaxInt, h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];
delta -=
intData[max(0,min(x+xMinInt, h))*(w+1)
+ max(1,min(y+yMaxInt+1, w))];
delta +=
intData[max(0,min(x+xMinInt, h))*(w+1)
+ max(0,min(y+yMaxInt  , w))];

delta *= (y+yMaxInt >= 1 and y+yMaxInt < w);
tmpArray[(x-1)*w + (y-1)] = delta;
}
}