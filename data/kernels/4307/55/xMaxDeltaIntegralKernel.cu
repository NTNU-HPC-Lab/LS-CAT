#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax) {

int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
const int y = id % w + 1; id /= w; // 1-indexed
const int x = id % h + 1; id /= h; // 1-indexed
const int & windowIdx = id;

if (windowIdx < nWindows and x <= h and y <= w) {

tmpArray += windowIdx * h * w;

// const int xMinInt = (int)ceil(xMin[windowIdx]-1);
const int yMinInt = (int)ceil(yMin[windowIdx]-1);
const int xMaxInt = (int)floor(xMax[windowIdx]);
const int yMaxInt = (int)floor(yMax[windowIdx]);

float delta = 0;

delta +=
intData[max(1,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMaxInt, w))];
delta -=
intData[max(1,min(x+xMaxInt+1, h))*(w+1)
+ max(0,min(y+yMinInt, w))];
delta +=
intData[max(0,min(x+xMaxInt  , h))*(w+1)
+ max(0,min(y+yMinInt, w))];

delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
tmpArray[(x-1)*w + (y-1)] = delta;
}
}