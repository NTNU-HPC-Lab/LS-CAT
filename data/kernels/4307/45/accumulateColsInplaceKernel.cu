#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulateColsInplaceKernel(float *input, int channels, int h, int w) {
// in-place.
// input is already a `channels * (h+1) x (w+1)` array

// global column index (of all `channels * w` columns in this image)
int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

if (colIdx < channels * w) {
input += (colIdx / w) * (h+1) * (w+1); // jump to current channel
colIdx %= w; // switch to local column index,
++colIdx;    // it's 1-indexed because first output column is always zero

input[colIdx] = 0; // first element of every column is always zero
double sum = 0;

for (int i = 1; i <= h; ++i) {
float *currentElement = &input[i * (w+1) + colIdx];
sum += static_cast<double>(*currentElement);
*currentElement = static_cast<float>(sum);
}
}
}