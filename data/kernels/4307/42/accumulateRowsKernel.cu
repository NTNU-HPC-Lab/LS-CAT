#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulateRowsKernel( float *input, float *output, int channels, int h, int w) {
// view multichannel image as a multiline single-channel image
int globalRowIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

if (globalRowIdx < channels * h) {
float *outputRow = output + (globalRowIdx + globalRowIdx / h + 1) * (w+1) + 1;
outputRow[-1] = 0;

double sum = 0;
for (int i = 0; i < w; ++i) {
sum += input[globalRowIdx * w + i];
outputRow[i] = static_cast<float>(sum);
}

// need to zero the (0,0) corner of the output separately >:(
output[(globalRowIdx / h) * (w+1) * (h+1)] = 0;
}
}