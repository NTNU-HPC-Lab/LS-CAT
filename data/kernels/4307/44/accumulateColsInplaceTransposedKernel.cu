#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulateColsInplaceTransposedKernel(float *input, int channels, int h, int w) {
// in-place.
// input is a `(w+1) x channels * (h+1)` array

// global column index (of all `channels * w` columns in this image)
int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

if (colIdx < channels * h) {
// need to zero the (0,0) corner of the output separately >:(
input[(colIdx / h) * (h+1)] = 0;

colIdx += colIdx / h + 1; // make `colIdx` the (h+1)-array indexer

input[colIdx] = 0; // first element of every column is always zero

double sum = 0;

for (int i = 1; i <= w; ++i) {
float *currentElement = &input[i * channels * (h+1) + colIdx];
sum += static_cast<double>(*currentElement);
*currentElement = static_cast<float>(sum);
}
}
}