#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixColour (float *a, float *b, int n){
int j= blockDim.x * blockIdx.x + threadIdx.x;

printf("Block = %d ; Thread = %d \n",blockIdx.x+1, threadIdx.x+1);

//	if(j<n){
//		for (int i=0; i<n; i++){
////			printf("Block = %d ; Thread = %d ; i = %d ; %f\n",blockIdx.x+1, j+1,i+1,b[i]);
//			if (a[j*n+i]==1){
//				if (b[j]==b[i]){
//					b[j]=-1;
//					break;
//				}
//			}
//		}
//	}

int *colour = new int[n];

memset(colour, 0, n*sizeof(int));

if (j<n){
for (int i=0; i<n; i++){
//printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
printf("Block = %d ; Thread = %d First For i = %d\n",blockIdx.x+1, threadIdx.x+1, i+1);
if (a[j*n+i]==1 && b[i]!=-1){
colour[(int)b[i]]=1;
}


//			if (i==j){
//				//atomicAdd(&b[i],1.0f);
//				b[i]+=1.0f;
//			}
}

for (int i=0; i<n; i++){

if (colour[i]==0){
printf("Block = %d ; Thread = %d Second For i = %d\n",blockIdx.x+1, threadIdx.x+1, i+1);
atomicAdd(&b[j],(float)i-b[j]);
break;
}
}


//		for (int i=0; i<n; i++){
//			printf("Third Block = %d ; ThreadId = %d ; Thread = %d ; i = %d ; %f\n",blockIdx.x+1, threadIdx.x+1, j+1,i+1,b[i]);
//		}

}




//	printf("I am thread no: %d from blocknumber: %d\n", threadIdx.x, blockIdx.x);

//b[j] = j+1;


}