#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomColouring (hiprandState* globalState, int *degreeCount, int n, int limit){

int i= blockDim.x * blockIdx.x + threadIdx.x;

hiprandState localState = globalState[i];
float RANDOM = hiprand_uniform( &localState );
globalState[i] = localState;

RANDOM *= (limit - 1 + 0.999999);
RANDOM += 1;

degreeCount[i] = (int) RANDOM;
}