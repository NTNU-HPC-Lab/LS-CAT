#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorValue (float *a, float *b, int n){
int i= blockDim.x * blockIdx.x + threadIdx.x;

if (i<n){
a[i]=threadIdx.x*2;
b[i]=threadIdx.x;
}

}