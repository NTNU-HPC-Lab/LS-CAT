#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conflictDetection (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict){

int i= blockDim.x * blockIdx.x + threadIdx.x;

if (i>=n){
return;
}

int myColour = degreeCount[i];

int start = -1, stop = -1;

start = vertexArray[i];


stop = vertexArray[i+1];

for (int j=start; j<stop; j++){
if (degreeCount[neighbourArray[j]-1] == myColour){

//			detectConflict[i]=1;
//			break;

if (i < neighbourArray[j]-1){
if (detectConflict[i]!=1){
detectConflict[i]=1;
}
}
else if (detectConflict[neighbourArray[j]-1]!=1){
detectConflict[neighbourArray[j]-1]=1;
}






//			if (detectConflict[i]!=1){
//				detectConflict[i]=1;
//			}
//
//			if (detectConflict[neighbourArray[j]-1]!=1){
//				detectConflict[neighbourArray[j]-1]=1;
//			}
}
}
}