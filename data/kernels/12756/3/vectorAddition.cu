#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddition (float *a, float *b, float *c, int n){
int i= blockDim.x * blockIdx.x + threadIdx.x;

if (i<n){
c[i] = a[i]+b[i];
}

}