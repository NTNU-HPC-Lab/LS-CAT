#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int sum = 1;  __global__ void degreeCalc (int *array){

int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i>=1000000){
return;
}

sum+=array[i];

//	if (i==999999){
//		printf("%d", sum);
//	}
}
__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){

int i= blockDim.x * blockIdx.x + threadIdx.x;

if (i>=n){
return;
}


int start = -1, stop = -1;
int diff=0;

start = vertexArray[i];

stop = vertexArray[i+1];


diff = stop-start;

degreeCount[i]=diff;
}