#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixColour (float *a, float *b, int n){
int j= blockDim.x * blockIdx.x + threadIdx.x;

if(j<n){
for (int i=0; i<n; i++){
printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
if (a[j*n+i]==1){
if (b[j]==b[i]){
b[j]=-1;
break;
}
}
}
}

//	int colour[10];
//
//	memset(colour, 0, 10*sizeof(float));

//	if (j<n){
//		for (int i=0; i<n; i++){
//			//printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
//			if (a[j*n+i]==1 && b[i]!=-1){
//				colour[(int)b[i]]=1;
//			}
//
//
////			if (i==j){
////				//atomicAdd(&b[i],1.0f);
////				b[i]+=1.0f;
////			}
//		}
//
//		for (int i=0; i<n; i++){
//			if (colour[i]==0){
//				b[j]=i;
//				break;
//			}
//
//
//
//		}
//
//
//		for (int i=0; i<n; i++){
//			printf("Thread = %d ; i = %d ; %f\n",j+1,i+1,b[i]);
//
//
//		}
//
//	}




//	printf("I am thread no: %d from blocknumber: %d\n", threadIdx.x, blockIdx.x);

//b[j] = j+1;


}