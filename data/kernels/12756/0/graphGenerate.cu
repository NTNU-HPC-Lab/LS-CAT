#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;




__global__ void graphGenerate (float *a, float *b, int n){
int i= blockDim.x * blockIdx.x + threadIdx.x;

if (i<n){
a[i]=threadIdx.x*2;
b[i]=threadIdx.x;
}

}