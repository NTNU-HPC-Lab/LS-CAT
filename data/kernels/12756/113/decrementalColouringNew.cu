#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decrementalColouringNew (int *vertexArray, int *neighbourArray, int n, int m, int *decrementalArray, int size){

int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i >= size){
return;
}



int startStart, startStop;
int me, you;
//	int otheri;
//	bool ipercent2 = false;

me = decrementalArray[i];

if (i%2 == 0){
you = decrementalArray[i+1];
//		otheri = i+1;
//		ipercent2 = true;
}
else{
you = decrementalArray[i-1];
//		otheri = i-1;
}

//printf("I am %d and I am deleting %d - %d\n", i, me, you);

startStart = vertexArray[me-1];

startStop = vertexArray[me];

for (int j=startStart; j<startStop; j++){
if (neighbourArray[j]==you){
neighbourArray[j]=0;
break;
}
}
}