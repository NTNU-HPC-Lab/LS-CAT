#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
const long long tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < n) {
p[tid] = v;
}
return;
}