#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
//int tid = blockIdx.x; //in N blocks
int tid = threadIdx.x; //in N threads
if(tid<N)
{
c[tid] = a[tid] + b[tid];
}
}