#include "hip/hip_runtime.h"
#include "includes.h"
__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx){
int x = threadIdx;
int y = blockIdx;
return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}
__global__ void mDivergence_TwoDim(float *div, float *u_dimX, float *u_dimY, float r_sStep) {
if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
int Left   = Idx - 1;
int Right  = Idx + 1;
int Top    = Idx + blockDim.x;
int Bottom = Idx - blockDim.x;

div[Idx] = ((u_dimX[Right]-u_dimX[Left])+(u_dimY[Top]-u_dimY[Bottom]))*r_sStep;
}