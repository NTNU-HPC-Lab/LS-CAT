#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IndexInteranlNode(bool *forest, int base, int step)
{
int left  = 2*(base+threadIdx.x);
int right = left + 1;
int offset = blockIdx.x*step;
forest[offset+base+threadIdx.x] = (forest[offset+left]&&forest[offset+right]);
}