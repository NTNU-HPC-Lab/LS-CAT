#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IndexLeafNode(const char *text, bool *forest, int text_size, int step)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int offset = blockIdx.x*step+blockDim.x;
forest[offset+threadIdx.x] = (text[idx] != '\n' && idx < text_size);
}