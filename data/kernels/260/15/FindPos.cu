#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindPos(int *pos, bool *forest, int text_size, int order, int step)
{
int text_idx = blockIdx.x * blockDim.x + threadIdx.x;
int offset = blockIdx.x*step;
if(text_idx < text_size) {
if(!forest[offset+blockDim.x+threadIdx.x]) {
pos[text_idx] = 0;
} else {
bool isCurBlock = true;
bool isLeftMost = (blockIdx.x < 1);
int  nodeIdx    = blockDim.x+threadIdx.x;
int  leftBound  = blockDim.x;
int  rightBound = 2*blockDim.x-1;
int  alignOrder = 0;
// bottom-up
while(alignOrder != order) {
int leftInx;
if(nodeIdx-1 < leftBound) {
if(isLeftMost) break;
isCurBlock = false;
leftInx = offset-step+rightBound;
} else {
leftInx = offset+nodeIdx-1;
}

if(!forest[leftInx]) break;

rightBound = leftBound-1;
leftBound /= 2;
nodeIdx /= 2;
alignOrder++;
}

// top-down
if(alignOrder == order && !isLeftMost) isCurBlock = false;
nodeIdx = (!isCurBlock)? rightBound
:(nodeIdx-1 < leftBound)? nodeIdx
:nodeIdx-1;

offset = offset - ((isCurBlock)? 0:step);
while(alignOrder != 0) {
if((alignOrder == order && isCurBlock) || forest[offset+2*nodeIdx+1]) {
nodeIdx = 2*nodeIdx;
} else {
nodeIdx = 2*nodeIdx+1;
}
alignOrder--;
}

pos[text_idx] = (isCurBlock)? (threadIdx.x-(nodeIdx-blockDim.x)+(forest[offset+nodeIdx]))
:(step-nodeIdx+threadIdx.x);
}
}
}