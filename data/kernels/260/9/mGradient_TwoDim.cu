#include "hip/hip_runtime.h"
#include "includes.h"
__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx){
int x = threadIdx;
int y = blockIdx;
return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}
__global__ void mGradient_TwoDim(float *u_dimX, float *u_dimY, float *scalar, float coeffX, float coeffY) {
if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
int Left   = Idx - 1;
int Right  = Idx + 1;
int Top    = Idx + blockDim.x;
int Bottom = Idx - blockDim.x;

u_dimX[Idx] -= (scalar[Right] - scalar[Left])*coeffX;
u_dimY[Idx] -= (scalar[Top] - scalar[Bottom])*coeffY;
}