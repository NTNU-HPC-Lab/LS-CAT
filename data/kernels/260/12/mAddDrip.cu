#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAddDrip(float *dense, int centerX, int centerY, float redius) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
int x = threadIdx.x;
int y = blockIdx.x;
float length = sqrt((float)((x-centerX)*(x-centerX))+(float)((y-centerY)*(y-centerY)));

if(length < redius) {
dense[Idx] += 200;
}
}