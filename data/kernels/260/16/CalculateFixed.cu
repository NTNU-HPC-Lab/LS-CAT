#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateFixed( const float *background, const float *target, const float *mask, float *fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox )
{
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt*yt+xt;
if (yt < ht and xt < wt and mask[curt] > 127.0f) {
bool nt_bnd = (yt == 0), wt_bnd = (xt == 0), st_bnd = (yt == ht-1), et_bnd = (xt == wt-1);
int North_t = (nt_bnd)? curt:(curt-wt);
int West_t  = (wt_bnd)? curt:(curt-1);
int South_t = (st_bnd)? curt:(curt+wt);
int East_t  = (et_bnd)? curt:(curt+1);

fixed[curt*3+0] = 4.0f*target[curt*3+0]-(target[North_t*3+0]+target[West_t*3+0]+target[South_t*3+0]+target[East_t*3+0]);
fixed[curt*3+1] = 4.0f*target[curt*3+1]-(target[North_t*3+1]+target[West_t*3+1]+target[South_t*3+1]+target[East_t*3+1]);
fixed[curt*3+2] = 4.0f*target[curt*3+2]-(target[North_t*3+2]+target[West_t*3+2]+target[South_t*3+2]+target[East_t*3+2]);

const int yb = oy+yt, xb = ox+xt;
const int curb = wb*yb+xb;
if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
bool nb_bnd = (yb == 0), wb_bnd = (xb == 0), sb_bnd = (yb == hb-1), eb_bnd = (xb == wb-1);
int North_b = (nb_bnd)? (curb):(curb-wb);
int West_b  = (wb_bnd)? (curb):(curb-1);
int South_b = (sb_bnd)? (curb):(curb+wb);
int East_b  = (eb_bnd)? (curb):(curb+1);

bool isMasked_n = (nt_bnd)? true:(mask[North_t] <= 127.0f);
bool isMasked_w = (wt_bnd)? true:(mask[West_t]  <= 127.0f);
bool isMasked_s = (st_bnd)? true:(mask[South_t] <= 127.0f);
bool isMasked_e = (et_bnd)? true:(mask[East_t]  <= 127.0f);

if(isMasked_n) {
fixed[curt*3+0] += background[North_b*3+0];
fixed[curt*3+1] += background[North_b*3+1];
fixed[curt*3+2] += background[North_b*3+2];
}

if(isMasked_w) {
fixed[curt*3+0] += background[West_b*3+0];
fixed[curt*3+1] += background[West_b*3+1];
fixed[curt*3+2] += background[West_b*3+2];
}

if(isMasked_s) {
fixed[curt*3+0] += background[South_b*3+0];
fixed[curt*3+1] += background[South_b*3+1];
fixed[curt*3+2] += background[South_b*3+2];
}

if(isMasked_e) {
fixed[curt*3+0] += background[East_b*3+0];
fixed[curt*3+1] += background[East_b*3+1];
fixed[curt*3+2] += background[East_b*3+2];
}
}
}
}