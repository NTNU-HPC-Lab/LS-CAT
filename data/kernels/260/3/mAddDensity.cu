#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAddDensity(float *dense, float *dense_old, float dt) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
dense[Idx] += dense_old[Idx]*dt;
}