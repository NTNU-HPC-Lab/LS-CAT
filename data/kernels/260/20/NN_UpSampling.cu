#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NN_UpSampling( float *target, const float *source, const int wt, const int ht )
{
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = y*wt+x;
const int curs = (y/2)*(wt/2)+x/2;
if(y < ht and x < wt) {
target[curt*3+0] = source[curs*3+0];
target[curt*3+1] = source[curs*3+1];
target[curt*3+2] = source[curs*3+2];
}
}