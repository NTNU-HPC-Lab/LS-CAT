#include "hip/hip_runtime.h"
#include "includes.h"
__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx){
int x = threadIdx;
int y = blockIdx;
return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}
__global__ void mSetFieldBoundary(float *field, float scalar) {
if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
int x = threadIdx.x;
int y = blockIdx.x;

if(x == 0 && y == 0) {
field[Idx] = field[Idx+blockDim.x+1]*scalar;
} else if(x == 0 && y == blockDim.x-1) {
field[Idx] = field[Idx-blockDim.x+1]*scalar;
} else if (x == blockDim.x-1 && y == 0) {
field[Idx] = field[Idx+blockDim.x-1]*scalar;
} else if (x == blockDim.x-1 && y == blockDim.x-1) {
field[Idx] = field[Idx-blockDim.x-1]*scalar;
} else if (x == 0) {
field[Idx] = field[Idx+1]*scalar;
} else if(x == blockDim.x-1) {
field[Idx] = field[Idx-1]*scalar;
} else if(y == 0) {
field[Idx] = field[Idx+blockDim.x]*scalar;
} else field[Idx] = field[Idx-blockDim.x]*scalar;
} else return;
}