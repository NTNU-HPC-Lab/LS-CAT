#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAddExternForce(float *w_dimX, float *w_dimY, float *f_dimX, float *f_dimY, float dt) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
w_dimX[Idx] = -0.5*w_dimX[Idx];
w_dimY[Idx] = -0.5*w_dimY[Idx];
}