#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAttachTexture(uint8_t *frame, float *dense) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
frame[Idx] = (dense[Idx] > 255.0)? 255:(uint8_t)(dense[Idx]);
}