#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mInitVelocity(float *u_dimX, float *u_dimY) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
u_dimX[Idx] = 0.f;
u_dimY[Idx] = 0.8f/(float)(blockIdx.x+1);
}