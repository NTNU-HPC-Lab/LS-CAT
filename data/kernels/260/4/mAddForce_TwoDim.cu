#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAddForce_TwoDim(float *velocityX, float *velocityY, float *forceX, float *forceY, float dt) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
velocityX[Idx] = (velocityX[Idx] >= 0.6)? velocityX[Idx]:velocityX[Idx]+forceX[Idx]*dt;
velocityY[Idx] = (velocityY[Idx] >= 0.6)? velocityY[Idx]:velocityY[Idx]+forceY[Idx]*dt;
}