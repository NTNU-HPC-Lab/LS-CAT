#include "hip/hip_runtime.h"
#include "includes.h"
__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx){
int x = threadIdx;
int y = blockIdx;
return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}
__global__ void mJocobi_TwoDim(float *x_new, float *x_old, float* b, float alpha, float rBeta) {
if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
int Left   = Idx - 1;
int Right  = Idx + 1;
int Top    = Idx + blockDim.x;
int Bottom = Idx - blockDim.x;

x_new[Idx] = ((x_old[Left]+x_old[Right]+x_old[Top]+x_old[Bottom])*alpha + b[Idx])*rBeta;
}