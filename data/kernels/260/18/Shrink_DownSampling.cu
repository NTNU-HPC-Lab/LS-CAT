#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Shrink_DownSampling( float *target, const float *source, const int wt, const int ht, const int ws, const int hs )
{
int y = blockIdx.y * blockDim.y + threadIdx.y;
int x = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = y*wt+x;
const int curs = (y*2)*ws+x*2;
if(y < ht and x < wt) {
target[curt*3+0] = (source[curs*3+0]+source[(curs+1)*3+0]+source[(curs+ws)*3+0]+source[(curs+ws+1)*3+0])/4.0f;
target[curt*3+1] = (source[curs*3+1]+source[(curs+1)*3+1]+source[(curs+ws)*3+1]+source[(curs+ws+1)*3+1])/4.0f;
target[curt*3+2] = (source[curs*3+2]+source[(curs+1)*3+2]+source[(curs+ws)*3+2]+source[(curs+ws+1)*3+2])/4.0f;
}
}