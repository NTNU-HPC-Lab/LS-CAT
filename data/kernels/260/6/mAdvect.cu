#include "hip/hip_runtime.h"
#include "includes.h"
__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx){
int x = threadIdx;
int y = blockIdx;
return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}
__global__ void mAdvect(float *new_data, float *old_data, float *xv, float *yv, float t_step, float s_stepX, float s_stepY) {
if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
float curr_x = (float)threadIdx.x;
float curr_y = (float)blockIdx.x;
float last_x = curr_x - t_step*s_stepX*xv[Idx];
float last_y = curr_y - t_step*s_stepY*yv[Idx];

if(last_x < 1.5)   last_x = 1.5;
if(last_x > 637.5) last_x = 637.5;
if(last_y < 1.5)   last_y = 1.5;
if(last_y > 477.5) last_y = 477.5;

// Bilinear Interpolation
float xDiff = last_x - (int)last_x;
float yDiff = last_y - (int)last_y;
int LeftTopX = (int)last_x;
int LeftTopY = (int)last_y;
int LeftTopIdx = LeftTopY * blockDim.x + LeftTopX;
new_data[Idx] = (xDiff*yDiff)*old_data[LeftTopIdx+blockDim.x+1]
+(xDiff*(1.f-yDiff))*old_data[LeftTopIdx+1]
+((1.f-xDiff)*yDiff)*old_data[LeftTopIdx+blockDim.x]
+((1.f-xDiff)*(1.f-yDiff))*old_data[LeftTopIdx];
}