#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PoissonImageCloningIteration( const float *fixed, const float *mask, const float *buf1, float *buf2, const int wt, const int ht )
{
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt*yt+xt;
if (yt < ht and xt < wt and mask[curt] > 127.0f) {
bool nt_bnd = (yt == 0), wt_bnd = (xt == 0), st_bnd = (yt == ht-1), et_bnd = (xt == wt-1);
int North_t = (nt_bnd)? curt:(curt-wt);
int West_t  = (wt_bnd)? curt:(curt-1);
int South_t = (st_bnd)? curt:(curt+wt);
int East_t  = (et_bnd)? curt:(curt+1);

bool isMasked_n = (nt_bnd)? true:(mask[North_t] <= 127.0f);
bool isMasked_w = (wt_bnd)? true:(mask[West_t]  <= 127.0f);
bool isMasked_s = (st_bnd)? true:(mask[South_t] <= 127.0f);
bool isMasked_e = (et_bnd)? true:(mask[East_t]  <= 127.0f);

buf2[curt*3+0] = fixed[curt*3+0];
buf2[curt*3+1] = fixed[curt*3+1];
buf2[curt*3+2] = fixed[curt*3+2];

if(!isMasked_n) {
buf2[curt*3+0] += buf1[North_t*3+0];
buf2[curt*3+1] += buf1[North_t*3+1];
buf2[curt*3+2] += buf1[North_t*3+2];
}

if(!isMasked_w) {
buf2[curt*3+0] += buf1[West_t*3+0];
buf2[curt*3+1] += buf1[West_t*3+1];
buf2[curt*3+2] += buf1[West_t*3+2];
}

if(!isMasked_s) {
buf2[curt*3+0] += buf1[South_t*3+0];
buf2[curt*3+1] += buf1[South_t*3+1];
buf2[curt*3+2] += buf1[South_t*3+2];
}

if(!isMasked_e) {
buf2[curt*3+0] += buf1[East_t*3+0];
buf2[curt*3+1] += buf1[East_t*3+1];
buf2[curt*3+2] += buf1[East_t*3+2];
}

buf2[curt*3+0] *= 0.25f;
buf2[curt*3+1] *= 0.25f;
buf2[curt*3+2] *= 0.25f;
}
}