#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mInitForce(float *f_dimX, float *f_dimY) {
int Idx = blockIdx.x * blockDim.x + threadIdx.x;
float x = (float)threadIdx.x;
float y = (float)blockIdx.x;
float length = sqrt((float)((x-320)*(x-320))+(float)((y-240)*(y-240)));

if(length < SWIRL_RADIUS) {
f_dimX[Idx] = (240.0-y)/length;
f_dimY[Idx] = (x-320.0)/length;
} else {
f_dimX[Idx] = f_dimY[Idx] = 0.f;
}
}