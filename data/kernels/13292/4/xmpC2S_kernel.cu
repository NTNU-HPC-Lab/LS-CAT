#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xmpC2S_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
//outer dimension = N
//inner dimension = limbs

//read strided in inner dimension`
//write coalesced in outer dimension
for(uint32_t i=blockIdx.x*blockDim.x+threadIdx.x;i<N;i+=blockDim.x*gridDim.x) {
for(uint32_t j=blockIdx.y*blockDim.y+threadIdx.y;j<limbs;j+=blockDim.y*gridDim.y) {
out[j*stride + i] = in[i*limbs + j];
}
}
}