#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xmpS2C_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
//outer dimension = limbs
//inner dimension = N

//read strided in inner dimension
//write coalesced in outer dimension
for(uint32_t i=blockIdx.x*blockDim.x+threadIdx.x;i<limbs;i+=blockDim.x*gridDim.x) {
for(uint32_t j=blockIdx.y*blockDim.y+threadIdx.y;j<N;j+=blockDim.y*gridDim.y) {
out[j*limbs + i] = in[i*stride + j];
}
}
}