#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void ReduceWKernelSimple(const uint8_t *src, float *dst, int width, int height)
{
int y = blockIdx.x * blockDim.x + threadIdx.x;
int x = blockIdx.y * 128;

if (y < height) {
float sum = 0;
for (int xend = min(x + 128, width); x < xend; ++x) {
sum += src[x + y * width];
}
atomicAdd(&dst[y], sum);
}
}