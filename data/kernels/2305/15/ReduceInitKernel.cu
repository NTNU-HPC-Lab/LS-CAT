#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void ReduceInitKernel(float *dst, int length)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;

if (x < length) {
dst[x] = 0;
}
}