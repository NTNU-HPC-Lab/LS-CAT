#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void TransposeKernelSimple(const uint8_t *src, uint8_t *dst, int width, int height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < width && y < height)
dst[y + x * height] = src[x + y * width];
}