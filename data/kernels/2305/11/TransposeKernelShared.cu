#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void TransposeKernelShared(const uint8_t *src, uint8_t *dst, int width, int height)
{
int tx = threadIdx.x;
int ty = threadIdx.y;
int xbase = blockIdx.x * blockDim.x;
int ybase = blockIdx.y * blockDim.y;

__shared__ uint8_t sbuf[16][16];

{
int x = xbase + tx;
int y = ybase + ty;
if (x < width && y < height)
sbuf[ty][tx] = src[x + y * width];
}

__syncthreads();

{
int x = xbase + ty;
int y = ybase + tx;
if (x < width && y < height)
dst[y + x * height] = sbuf[tx][ty];
}
}