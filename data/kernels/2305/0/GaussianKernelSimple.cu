#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void GaussianKernelSimple(const uint8_t *src, uint8_t *dst, int width, int height, int step)
{
const float filter[5][5] = {
{ 0.002969017f, 0.01330621f, 0.021938231f, 0.01330621f, 0.002969017f },
{ 0.01330621f, 0.059634295f, 0.098320331f, 0.059634295f, 0.01330621f },
{ 0.021938231f, 0.098320331f, 0.162102822f, 0.098320331f, 0.021938231f },
{ 0.01330621f, 0.059634295f, 0.098320331f, 0.059634295f, 0.01330621f },
{ 0.002969017f, 0.01330621f, 0.021938231f, 0.01330621f, 0.002969017f },
};

int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < width && y < height) {
float sum = 0;
for (int dy = 0; dy < 5; ++dy) {
for (int dx = 0; dx < 5; ++dx) {
sum += filter[dy][dx] * src[(x + dx) + (y + dy) * step];
}
}
dst[x + y * step] = (int)(sum + 0.5f);
}
}