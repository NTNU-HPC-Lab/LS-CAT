#include "hip/hip_runtime.h"
#include "includes.h"



__device__ float ReduceFunc(int tid, float* buf)
{
if (tid < 256) {
buf[tid] += buf[tid + 256];
}
__syncthreads();
if (tid < 128) {
buf[tid] += buf[tid + 128];
}
__syncthreads();
if (tid < 64) {
buf[tid] += buf[tid + 64];
}
__syncthreads();
float sum;
if (tid < 32) {
sum = buf[tid] + buf[tid + 32];
sum += __shfl_down_sync(0xffffffff, sum, 16);
sum += __shfl_down_sync(0xffffffff, sum, 8);
sum += __shfl_down_sync(0xffffffff, sum, 4);
sum += __shfl_down_sync(0xffffffff, sum, 2);
sum += __shfl_down_sync(0xffffffff, sum, 1);
}
return sum;
}
__global__ void ReduceWKernelFast(const uint8_t *src, float *dst, int width, int height)
{
int tid = threadIdx.x;
int y = blockIdx.y;

__shared__ float sbuf[512];

float sum = 0;
for (int x = tid; x < width; x += 512) {
sum += src[x + y * width];
}

sbuf[tid] = sum;
__syncthreads();

sum = ReduceFunc(tid, sbuf);

if (tid == 0)
dst[y] = sum;
}