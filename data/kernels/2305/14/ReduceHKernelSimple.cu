#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void ReduceHKernelSimple(const uint8_t *src, float *dst, int width, int height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;

if (x < width) {
float sum = 0;
for (int y = 0; y < height; ++y) {
sum += src[x + y * width];
}
dst[x] = sum;
}
}