#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void TransposeKernelFast2(const uint8_t *src, uint8_t *dst, int width, int height)
{
int tx = threadIdx.x;
int ty = threadIdx.y;
int xbase = blockIdx.x * 32;
int ybase = blockIdx.y * 32;

__shared__ uint8_t sbuf[32][32+4];

{
int x = xbase + tx;
if (x < width) {
int yend = min(ybase + 32, height);
for (int tyy = ty, y = ybase + ty; y < yend; tyy += 8, y += 8) {
sbuf[tyy][tx] = src[x + y * width];
}
}
}

__syncthreads();

{
int y = ybase + tx;
if (y < height) {
int xend = min(xbase + 32, width);
for (int tyy = ty, x = xbase + ty; x < xend; tyy += 8, x += 8) {
dst[y + x * height] = sbuf[tx][tyy];
}
}
}
}