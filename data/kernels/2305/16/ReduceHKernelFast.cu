#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void ReduceHKernelFast(const uint8_t *src, float *dst, int width, int height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * 128;

if (x < width) {
float sum = 0;
for (int yend = min(y + 128, height); y < yend; ++y) {
sum += src[x + y * width];
}
atomicAdd(&dst[x], sum);
}
}