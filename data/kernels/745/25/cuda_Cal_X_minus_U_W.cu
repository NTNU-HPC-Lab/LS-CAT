#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_Cal_X_minus_U_W(float *Y, float *U, float *X, int *Weight, int nRows, int nCols) {
unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y, index;

float X_temp, U_temp, Y_temp;

if ((Tidx < nCols) && (Tidy < nRows)) {
index = Tidx + Tidy * nCols;

X_temp = (X[index] / (nRows * nCols));
U_temp = U[index];

Y_temp = (1 - Weight[index]) * (X_temp + U_temp);

Y[index] = Y_temp;
U[index] = U_temp + X_temp - Y_temp;
}
}