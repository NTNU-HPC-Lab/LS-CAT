#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_Pad_Dict(float *PadD, float *D, int nRows_D, int nCols_D, int nFilts, int nRows, int nCols) {
unsigned int Tidx_D = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int Tidy_D = threadIdx.y + blockIdx.y * blockDim.y;

int Dim_D = nRows_D * nFilts;
int i,j;

if ((Tidx_D < nCols_D) && (Tidy_D < nRows_D)) {

for ( i = Tidy_D, j = Tidy_D ; i < Dim_D; i += nRows_D, j += nRows)
PadD[Tidx_D + j * nCols] = D[Tidx_D + i * nCols_D];
}
}