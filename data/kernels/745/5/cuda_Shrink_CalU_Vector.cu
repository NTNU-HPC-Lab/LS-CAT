#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_Shrink_CalU_Vector(float *Y, float *U, float *X, float lambda, float *L1Weight, int nRows, int nCols, int nFilts) {
unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y, index;

float WLambda;
float absxV1, X_temp, U_temp, Y_temp;

if ((Tidx < nCols) && (Tidy < nRows)) {

for (int k = 0; k < nFilts; k += 1) {
index = Tidx + (Tidy + nRows * k) * nCols;

X_temp = (X[index] / (nRows * nCols));
U_temp = U[index];

WLambda = lambda * L1Weight[k];

Y_temp = X_temp + U_temp;
absxV1 = fabs(Y_temp) - WLambda;

Y_temp = signbit(-absxV1) * copysign(absxV1, Y_temp);

Y[index] = Y_temp;
U[index] = U_temp + X_temp - Y_temp;
}
}
}