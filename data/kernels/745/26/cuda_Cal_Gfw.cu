#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_Cal_Gfw(float *GfW, float2 *Grf, float2 *Gcf, int nRows, int nCols) {
unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int index;

float GfW_temp;
float2 Grf_temp, Gcf_temp;

if ((Tidx < nCols) && (Tidy < nRows)) {

index = Tidx + Tidy * nCols;

Grf_temp = Grf[index];
Gcf_temp = Gcf[index];
GfW_temp = Grf_temp.x * Grf_temp.x + Grf_temp.y * Grf_temp.y +
Gcf_temp.x * Gcf_temp.x + Gcf_temp.y * Gcf_temp.y;

GfW[index] = GfW_temp;
}
}