#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose(int N, double *A)
{
int row,col,k;
double temp;
k = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x+threadIdx.x);
row = k/N;
col = k - row*N;
if(row<col){
temp = A[row*N+col];
A[row*N+col] = A[col*N+row];
A[col*N+row] = temp;
}
}