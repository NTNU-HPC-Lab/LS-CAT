#include "hip/hip_runtime.h"
#include "includes.h"
// Template for Assignment 1: CUDA
// Use "icc -O -openmp" to compile

#define threshold 1e-4
#define n (2048)
void init(void);
void ref(void);
void test(void);
void compare(int N, double *wref, double *w);


__global__ void test_kernel(int N, double *A, double *B, double *X)
{
int i,j,k;
double temp;
// Template version uses only one thread, which does all the work
// This must be changed (and the launch parameters) to exploit GPU parallelism
// You can make any changes; only requirement is that correctness test passes
k = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x+threadIdx.x);
//if(threadIdx.x == 0) {
//for(k=0;k<n;k++){
/*
if(k<n){
for (i=0;i<n;i++){
temp = B[k*N+i]; // temp = b[k][i];
for (j=0;j<i;j++) temp = temp - A[i*N+j] * X[k*N+j]; // temp = temp - a[i][j]*x[k][j];
X[k*N+i] = temp/A[i*N+i]; //x[k][i] = temp/a[i][i];
}
}
*/
if(k<n){
for (i=0;i<n;i++){
temp = B[i*N+k]; // temp = b[k][i];
for (j=0;j<i;j++) temp = temp - A[j*N+i] * X[j*N+k]; // temp = temp - a[i][j]*x[k][j];
X[i*N+k] = temp/A[i*N+i]; //x[k][i] = temp/a[i][i];
}
}
//  }
// }
}