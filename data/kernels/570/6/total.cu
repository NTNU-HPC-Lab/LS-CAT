#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void total(float *input, float *output, int len)
{
//@@ Load a segment of the input vector into shared memory
//@@ Traverse the reduction tree
//@@ Write the computed sum of the block to the output vector at the
//@@ correct index
__shared__ float partSum[2 * BLOCK_SIZE];
unsigned int tx = threadIdx.x;
unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;
//Loading input floats to shared memory
//Take care of the boundary conditions
if (start + tx < len){
partSum[tx] = input[start + tx];
if (start + BLOCK_SIZE + tx <len) partSum[BLOCK_SIZE + tx] = input[start + BLOCK_SIZE + tx];
else partSum[BLOCK_SIZE + tx] = 0;
}
else{
partSum[tx] = 0;
partSum[BLOCK_SIZE + tx] = 0;
}
unsigned int stride;
for (stride = BLOCK_SIZE; stride > 0; stride = stride / 2){
__syncthreads();
if (tx < stride) partSum[tx] += partSum[tx + stride];
}
if (tx == 0){ output[blockIdx.x] = partSum[0]; }
}