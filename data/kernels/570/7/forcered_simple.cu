#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void forcered_simple(float * force, float * forcered){
int index = threadIdx.x + blockDim.x*blockIdx.x;
int i = 0;
int findex;
__shared__ float forcered_sh[3 * N_ATOMS];
//if (index == 0){ printf("In force reduction kernel! \n"); }
if (index < 3 * N_ATOMS){
forcered_sh[index] = 0.0f;
}
__syncthreads();
if (index < 3 * N_ATOMS){
findex = int(index / N_ATOMS)*N_ATOMS*N_ATOMS + index % N_ATOMS;
for (i = 0; i < N_ATOMS; i++){
forcered_sh[index] += force[findex + i*N_ATOMS];
}
}
__syncthreads();
if (index < 3 * N_ATOMS){
forcered[index] = forcered_sh[index];
}
/*if (index == 0){
printf("forcered [0]= %f \n", forcered[0]);
printf("forcered [2]= %f \n", forcered[2]);
printf("forcered [4]= %f \n \n", forcered[4]);
}*/
}