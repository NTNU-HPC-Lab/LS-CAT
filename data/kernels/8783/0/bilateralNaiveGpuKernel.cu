#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCKDIM = 16;

/**
* @brief      Calculates the Euclidean distance between two points (x0, y0) and
*             (x1, y1)
*
* @param[in]  x0    The x0 coordinate
* @param[in]  y0    The y0 coordinate
* @param[in]  x1    The x1 coordinate
* @param[in]  y1    The y1 coordinate
*
* @return     The distance between the two points
*/
__device__ inline float gaussian(float x, float mu, float sigma)
{
return static_cast<float>(expf(-((x - mu) * (x - mu))/(2 * sigma * sigma)) / (2 * M_PI * sigma * sigma));
}
__device__ inline float distance(int x0, int y0, int x1, int y1)
{
return static_cast<float>(sqrtf( (x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1) ));
}
__global__ void bilateralNaiveGpuKernel( float* inputImage, float* outputImage, int rows, int cols, uint32_t window, float sigmaD, float sigmaR)
{
float filteredPixel, neighbourPixel, currentPixel;
float wP, gR, gD;
int neighbourCol;
int neighbourRow;

const int col = blockIdx.x * blockDim.x + threadIdx.x;
const int row = blockIdx.y * blockDim.y + threadIdx.y;

if (col >= cols || row >= rows)
{
return;
}

filteredPixel = 0;
wP = 0;

for (int windowCol = 0; windowCol < window; windowCol++)
{
for (int windowRow = 0; windowRow < window; windowRow++)
{
neighbourCol = col - (window / 2) - windowCol;
neighbourRow = row - (window / 2) - windowRow;

// Prevent us indexing into regions that don't exist
if (neighbourCol < 0)
{
neighbourCol = 0;
}
if (neighbourRow < 0)
{
neighbourRow = 0;
}

neighbourPixel = inputImage[neighbourCol + neighbourRow * cols];
currentPixel = inputImage[col + row * cols];

// Intensity factor
gR = gaussian(neighbourPixel - currentPixel, 0.0, sigmaR);
// Distance factor
gD = gaussian(distance(col, row, neighbourCol, neighbourRow), 0.0, sigmaD);

filteredPixel += neighbourPixel * (gR * gD);

wP += (gR * gD);
}
}
outputImage[col + row * cols] = filteredPixel / wP;
}