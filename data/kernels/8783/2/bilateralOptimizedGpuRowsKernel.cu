#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCKDIM = 16;

/**
* @brief      Calculates the Euclidean distance between two points (x0, y0) and
*             (x1, y1)
*
* @param[in]  x0    The x0 coordinate
* @param[in]  y0    The y0 coordinate
* @param[in]  x1    The x1 coordinate
* @param[in]  y1    The y1 coordinate
*
* @return     The distance between the two points
*/
__device__ inline float gaussian(float x, float mu, float sigma)
{
return static_cast<float>(expf(-((x - mu) * (x - mu))/(2 * sigma * sigma)) / (2 * M_PI * sigma * sigma));
}
__device__ inline float distance(int x0, int y0, int x1, int y1)
{
return static_cast<float>(sqrtf( (x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1) ));
}
__global__ void bilateralOptimizedGpuRowsKernel( float* inputImage, float* outputImage, int rows, int cols, uint32_t window, float sigmaD, float sigmaR)
{
float filteredPixel, neighbourPixel, currentPixel;
float wP, gR, gD;
int neighbourRow;

const int col = blockIdx.x * blockDim.x + threadIdx.x;
const int row = blockIdx.y * blockDim.y + threadIdx.y;

if (col >= cols || row >= rows)
{
return;
}

filteredPixel = 0;
wP = 0;

#pragma unroll
for (int windowRow = 0; windowRow < window; windowRow++)
{
neighbourRow = row - (window / 2) - windowRow;

// Prevent us indexing into regions that don't exist
if (neighbourRow < 0)
{
neighbourRow = 0;
}

neighbourPixel = inputImage[col + neighbourRow * cols];
currentPixel = inputImage[col + row * cols];

// Intensity factor
gR = gaussian(neighbourPixel - currentPixel, 0.0, sigmaR);
// Distance factor
gD = gaussian(distance(col, row, col, neighbourRow), 0.0, sigmaD);

filteredPixel += neighbourPixel * (gR * gD);

wP += (gR * gD);
}

outputImage[col + row * cols] = filteredPixel / wP;
}