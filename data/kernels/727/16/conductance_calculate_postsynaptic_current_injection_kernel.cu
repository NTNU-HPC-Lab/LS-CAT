#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conductance_calculate_postsynaptic_current_injection_kernel( float* decay_term_values, float* reversal_values, int num_decay_terms, int* synapse_decay_values, float* neuron_wise_conductance_traces, float* d_neurons_current_injections, float * d_membrane_potentials_v, float timestep, size_t total_number_of_neurons){

int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {

float membrane_potential_v = d_membrane_potentials_v[idx];

for (int decay_id = 0; decay_id < num_decay_terms; decay_id++){
if (decay_id == 0)
d_neurons_current_injections[idx] = 0.0f;
float synaptic_conductance_g = neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons];
// First decay the conductance values as required
synaptic_conductance_g *= expf(- timestep / decay_term_values[decay_id]);
neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons] = synaptic_conductance_g;
d_neurons_current_injections[idx] += synaptic_conductance_g * (reversal_values[decay_id] - membrane_potential_v);
}

idx += blockDim.x * gridDim.x;

}
}