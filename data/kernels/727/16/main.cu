#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "conductance_calculate_postsynaptic_current_injection_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *decay_term_values = NULL;
hipMalloc(&decay_term_values, XSIZE*YSIZE);
float *reversal_values = NULL;
hipMalloc(&reversal_values, XSIZE*YSIZE);
int num_decay_terms = 1;
int *synapse_decay_values = NULL;
hipMalloc(&synapse_decay_values, XSIZE*YSIZE);
float *neuron_wise_conductance_traces = NULL;
hipMalloc(&neuron_wise_conductance_traces, XSIZE*YSIZE);
float *d_neurons_current_injections = NULL;
hipMalloc(&d_neurons_current_injections, XSIZE*YSIZE);
float *d_membrane_potentials_v = NULL;
hipMalloc(&d_membrane_potentials_v, XSIZE*YSIZE);
float timestep = 1;
size_t total_number_of_neurons = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(decay_term_values,reversal_values,num_decay_terms,synapse_decay_values,neuron_wise_conductance_traces,d_neurons_current_injections,d_membrane_potentials_v,timestep,total_number_of_neurons);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(decay_term_values,reversal_values,num_decay_terms,synapse_decay_values,neuron_wise_conductance_traces,d_neurons_current_injections,d_membrane_potentials_v,timestep,total_number_of_neurons);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
conductance_calculate_postsynaptic_current_injection_kernel<<<gridBlock,threadBlock>>>(decay_term_values,reversal_values,num_decay_terms,synapse_decay_values,neuron_wise_conductance_traces,d_neurons_current_injections,d_membrane_potentials_v,timestep,total_number_of_neurons);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}