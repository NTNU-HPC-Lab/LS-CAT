#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "izhikevich_update_membrane_potentials_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_membrane_potentials_v = NULL;
hipMalloc(&d_membrane_potentials_v, XSIZE*YSIZE);
float *d_states_u = NULL;
hipMalloc(&d_states_u, XSIZE*YSIZE);
float *d_param_a = NULL;
hipMalloc(&d_param_a, XSIZE*YSIZE);
float *d_param_b = NULL;
hipMalloc(&d_param_b, XSIZE*YSIZE);
float *d_current_injections = NULL;
hipMalloc(&d_current_injections, XSIZE*YSIZE);
float *thresholds_for_action_potentials = NULL;
hipMalloc(&thresholds_for_action_potentials, XSIZE*YSIZE);
float *last_spike_time_of_each_neuron = NULL;
hipMalloc(&last_spike_time_of_each_neuron, XSIZE*YSIZE);
float *resting_potentials = NULL;
hipMalloc(&resting_potentials, XSIZE*YSIZE);
float current_time_in_seconds = 1;
float timestep = 1;
size_t total_number_of_neurons = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
izhikevich_update_membrane_potentials_kernel<<<gridBlock,threadBlock>>>(d_membrane_potentials_v,d_states_u,d_param_a,d_param_b,d_current_injections,thresholds_for_action_potentials,last_spike_time_of_each_neuron,resting_potentials,current_time_in_seconds,timestep,total_number_of_neurons);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
izhikevich_update_membrane_potentials_kernel<<<gridBlock,threadBlock>>>(d_membrane_potentials_v,d_states_u,d_param_a,d_param_b,d_current_injections,thresholds_for_action_potentials,last_spike_time_of_each_neuron,resting_potentials,current_time_in_seconds,timestep,total_number_of_neurons);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
izhikevich_update_membrane_potentials_kernel<<<gridBlock,threadBlock>>>(d_membrane_potentials_v,d_states_u,d_param_a,d_param_b,d_current_injections,thresholds_for_action_potentials,last_spike_time_of_each_neuron,resting_potentials,current_time_in_seconds,timestep,total_number_of_neurons);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}