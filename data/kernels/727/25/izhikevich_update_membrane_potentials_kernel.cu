#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void izhikevich_update_membrane_potentials_kernel(float *d_membrane_potentials_v, float *d_states_u, float *d_param_a, float *d_param_b, float* d_current_injections, float* thresholds_for_action_potentials, float* last_spike_time_of_each_neuron, float* resting_potentials, float current_time_in_seconds, float timestep, size_t total_number_of_neurons) {

// We require the equation timestep in ms:
float eqtimestep = timestep*1000.0f;
// Get thread IDs
int idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < total_number_of_neurons) {
// Update the neuron states according to the Izhikevich equations
float v_update = 0.04f*d_membrane_potentials_v[idx]*d_membrane_potentials_v[idx]
+ 5.0f*d_membrane_potentials_v[idx]
+ 140
- d_states_u[idx]
+ d_current_injections[idx];

d_membrane_potentials_v[idx] += eqtimestep*v_update;
d_states_u[idx] += eqtimestep*(d_param_a[idx] * (d_param_b[idx] * d_membrane_potentials_v[idx] -
d_states_u[idx]));

if (d_membrane_potentials_v[idx] >= thresholds_for_action_potentials[idx]){
d_membrane_potentials_v[idx] = resting_potentials[idx];
last_spike_time_of_each_neuron[idx] = current_time_in_seconds;
}

idx += blockDim.x * gridDim.x;
}
__syncthreads();
}