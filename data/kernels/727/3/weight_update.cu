#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weight_update( int* postsyn_neuron, bool* neuron_in_plasticity_set, float* current_weight, float* weight_divisor, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses){

// Global Index
int indx = threadIdx.x + blockIdx.x * blockDim.x;

while (indx < total_number_of_plastic_synapses) {
int idx = d_plastic_synapse_indices[indx];
int postneuron = postsyn_neuron[idx];
if (neuron_in_plasticity_set[postneuron]){
float division_value = weight_divisor[postneuron];
//if (division_value != 1.0)
//printf("%f, %f, %f wat \n", division_value, current_weight[idx], (current_weight[idx] / division_value));
if (division_value != 1.0)
current_weight[idx] /= division_value;
}
indx += blockDim.x * gridDim.x;
}
}