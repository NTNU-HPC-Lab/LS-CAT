#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "weight_update.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *postsyn_neuron = NULL;
hipMalloc(&postsyn_neuron, XSIZE*YSIZE);
bool *neuron_in_plasticity_set = NULL;
hipMalloc(&neuron_in_plasticity_set, XSIZE*YSIZE);
float *current_weight = NULL;
hipMalloc(&current_weight, XSIZE*YSIZE);
float *weight_divisor = NULL;
hipMalloc(&weight_divisor, XSIZE*YSIZE);
int *d_plastic_synapse_indices = NULL;
hipMalloc(&d_plastic_synapse_indices, XSIZE*YSIZE);
size_t total_number_of_plastic_synapses = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
weight_update<<<gridBlock,threadBlock>>>(postsyn_neuron,neuron_in_plasticity_set,current_weight,weight_divisor,d_plastic_synapse_indices,total_number_of_plastic_synapses);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
weight_update<<<gridBlock,threadBlock>>>(postsyn_neuron,neuron_in_plasticity_set,current_weight,weight_divisor,d_plastic_synapse_indices,total_number_of_plastic_synapses);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
weight_update<<<gridBlock,threadBlock>>>(postsyn_neuron,neuron_in_plasticity_set,current_weight,weight_divisor,d_plastic_synapse_indices,total_number_of_plastic_synapses);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}