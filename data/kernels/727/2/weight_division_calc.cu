#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weight_division_calc( float* sum_squared_afferent_values, float* afferent_weight_change_updater, float* weight_divisor, bool* neuron_in_plasticity_set, size_t total_number_of_neurons)
{
// Global Index
int idx = threadIdx.x + blockIdx.x * blockDim.x;

while (idx < total_number_of_neurons) {
if (neuron_in_plasticity_set[idx])
{
if ((sum_squared_afferent_values[idx] - afferent_weight_change_updater[idx] < 0.01))
printf("NORMALIZATION DIFF VERY LARGE. DANGER OF SYNAPSES ALL -> ZERO");
weight_divisor[idx] = sqrtf(sum_squared_afferent_values[idx] + afferent_weight_change_updater[idx]) / sqrtf(sum_squared_afferent_values[idx]);
}
idx += blockDim.x * gridDim.x;
}
__syncthreads();
}