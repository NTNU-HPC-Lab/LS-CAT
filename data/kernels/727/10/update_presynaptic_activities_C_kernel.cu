#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_presynaptic_activities_C_kernel (float* d_recent_presynaptic_activities_C, float* d_time_of_last_spike_to_reach_synapse, float timestep, float current_time_in_seconds, float synaptic_neurotransmitter_concentration_alpha_C, float decay_term_tau_C, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses) {

int indx = threadIdx.x + blockIdx.x * blockDim.x;
while (indx < total_number_of_plastic_synapses) {
int idx = d_plastic_synapse_indices[indx];

float recent_presynaptic_activity_C = d_recent_presynaptic_activities_C[idx];

float new_recent_presynaptic_activity_C = (1 - (timestep/decay_term_tau_C)) * recent_presynaptic_activity_C;

if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
new_recent_presynaptic_activity_C += timestep * synaptic_neurotransmitter_concentration_alpha_C * (1 - recent_presynaptic_activity_C);
}

if (recent_presynaptic_activity_C != new_recent_presynaptic_activity_C) {
d_recent_presynaptic_activities_C[idx] = new_recent_presynaptic_activity_C;
}

indx += blockDim.x * gridDim.x;

}

}