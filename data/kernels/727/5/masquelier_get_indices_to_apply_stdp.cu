#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void masquelier_get_indices_to_apply_stdp (int* d_postsyns, float* d_last_spike_time_of_each_neuron, float* d_time_of_last_spike_to_reach_synapse, int* d_index_of_last_afferent_synapse_to_spike, bool* d_isindexed_ltd_synapse_spike, int* d_index_of_first_synapse_spiked_after_postneuron, float currtime, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses){
int indx = threadIdx.x + blockIdx.x * blockDim.x;

// Running through all neurons:
while (indx < total_number_of_plastic_synapses){
int idx = d_plastic_synapse_indices[indx];
int postsynaptic_neuron = d_postsyns[idx];

// Check whether a synapse reached a neuron this timestep
if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
// Atomic Exchange the new synapse index
atomicExch(&d_index_of_last_afferent_synapse_to_spike[postsynaptic_neuron], idx);
}

// Check (if we need to) whether a synapse has fired
if (!d_isindexed_ltd_synapse_spike[postsynaptic_neuron]){
if (d_time_of_last_spike_to_reach_synapse[idx] == currtime){
d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = true;
atomicExch(&d_index_of_first_synapse_spiked_after_postneuron[postsynaptic_neuron], idx);
}
}
// Increment index
indx += blockDim.x * gridDim.x;
}
}