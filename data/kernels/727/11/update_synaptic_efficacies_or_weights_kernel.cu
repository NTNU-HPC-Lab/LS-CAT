#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_synaptic_efficacies_or_weights_kernel (float * d_recent_presynaptic_activities_C, float * d_recent_postsynaptic_activities_D, int* d_postsynaptic_neuron_indices, float* d_synaptic_efficacies_or_weights, float current_time_in_seconds, float * d_time_of_last_spike_to_reach_synapse, float * d_last_spike_time_of_each_neuron, float learning_rate_rho, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses) {

int indx = threadIdx.x + blockIdx.x * blockDim.x;

while (indx < total_number_of_plastic_synapses) {
int idx = d_plastic_synapse_indices[indx];

float synaptic_efficacy_delta_g = d_synaptic_efficacies_or_weights[idx];
float new_synaptic_efficacy = synaptic_efficacy_delta_g;

float new_componet = 0.0;

int postsynaptic_neuron_index = d_postsynaptic_neuron_indices[idx];

if (d_last_spike_time_of_each_neuron[postsynaptic_neuron_index] == current_time_in_seconds) {
float recent_presynaptic_activity_C = d_recent_presynaptic_activities_C[idx];
float new_componet_addition = ((1 - synaptic_efficacy_delta_g) * recent_presynaptic_activity_C);
new_componet += new_componet_addition;
}

if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
float recent_postsynaptic_activity_D = d_recent_postsynaptic_activities_D[postsynaptic_neuron_index];
new_componet -= (synaptic_efficacy_delta_g * recent_postsynaptic_activity_D);
}

if (new_componet != 0.0) {
new_componet = learning_rate_rho * new_componet;
new_synaptic_efficacy += new_componet;
}

if (synaptic_efficacy_delta_g != new_synaptic_efficacy) {
new_synaptic_efficacy = max(new_synaptic_efficacy, 0.0);
new_synaptic_efficacy = min(new_synaptic_efficacy, 1.0);

d_synaptic_efficacies_or_weights[idx] = new_synaptic_efficacy;
}



indx += blockDim.x * gridDim.x;
}
}