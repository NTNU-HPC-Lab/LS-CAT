#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conductance_move_spikes_towards_synapses_kernel( int* d_spikes_travelling_to_synapse, float current_time_in_seconds, int* circular_spikenum_buffer, int* spikeid_buffer, int bufferloc, int buffersize, int total_number_of_synapses, float* d_time_of_last_spike_to_reach_synapse, int* postsynaptic_neuron_indices, float * neuron_wise_conductance_trace, int * synaptic_decay_id, int total_number_of_neurons, float * d_synaptic_efficacies_or_weights, float * d_biological_conductance_scaling_constants_lambda, float timestep){

int indx = threadIdx.x + blockIdx.x * blockDim.x;
while (indx < circular_spikenum_buffer[bufferloc]) {
int idx = spikeid_buffer[bufferloc*total_number_of_synapses + indx];

// Update Synapses
d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
int postsynaptic_neuron_id = postsynaptic_neuron_indices[idx];
int trace_id = synaptic_decay_id[idx];
float synaptic_efficacy = d_biological_conductance_scaling_constants_lambda[idx] * d_synaptic_efficacies_or_weights[idx];
atomicAdd(&neuron_wise_conductance_trace[total_number_of_neurons*trace_id + postsynaptic_neuron_id], synaptic_efficacy);

indx += blockDim.x * gridDim.x;
}
__syncthreads();
}