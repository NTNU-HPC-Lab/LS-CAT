#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "conductance_move_spikes_towards_synapses_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *d_spikes_travelling_to_synapse = NULL;
hipMalloc(&d_spikes_travelling_to_synapse, XSIZE*YSIZE);
float current_time_in_seconds = 1;
int *circular_spikenum_buffer = NULL;
hipMalloc(&circular_spikenum_buffer, XSIZE*YSIZE);
int *spikeid_buffer = NULL;
hipMalloc(&spikeid_buffer, XSIZE*YSIZE);
int bufferloc = 1;
int buffersize = XSIZE*YSIZE;
int total_number_of_synapses = 1;
float *d_time_of_last_spike_to_reach_synapse = NULL;
hipMalloc(&d_time_of_last_spike_to_reach_synapse, XSIZE*YSIZE);
int *postsynaptic_neuron_indices = NULL;
hipMalloc(&postsynaptic_neuron_indices, XSIZE*YSIZE);
float *neuron_wise_conductance_trace = NULL;
hipMalloc(&neuron_wise_conductance_trace, XSIZE*YSIZE);
int *synaptic_decay_id = NULL;
hipMalloc(&synaptic_decay_id, XSIZE*YSIZE);
int total_number_of_neurons = 1;
float *d_synaptic_efficacies_or_weights = NULL;
hipMalloc(&d_synaptic_efficacies_or_weights, XSIZE*YSIZE);
float *d_biological_conductance_scaling_constants_lambda = NULL;
hipMalloc(&d_biological_conductance_scaling_constants_lambda, XSIZE*YSIZE);
float timestep = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
conductance_move_spikes_towards_synapses_kernel<<<gridBlock,threadBlock>>>(d_spikes_travelling_to_synapse,current_time_in_seconds,circular_spikenum_buffer,spikeid_buffer,bufferloc,buffersize,total_number_of_synapses,d_time_of_last_spike_to_reach_synapse,postsynaptic_neuron_indices,neuron_wise_conductance_trace,synaptic_decay_id,total_number_of_neurons,d_synaptic_efficacies_or_weights,d_biological_conductance_scaling_constants_lambda,timestep);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
conductance_move_spikes_towards_synapses_kernel<<<gridBlock,threadBlock>>>(d_spikes_travelling_to_synapse,current_time_in_seconds,circular_spikenum_buffer,spikeid_buffer,bufferloc,buffersize,total_number_of_synapses,d_time_of_last_spike_to_reach_synapse,postsynaptic_neuron_indices,neuron_wise_conductance_trace,synaptic_decay_id,total_number_of_neurons,d_synaptic_efficacies_or_weights,d_biological_conductance_scaling_constants_lambda,timestep);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
conductance_move_spikes_towards_synapses_kernel<<<gridBlock,threadBlock>>>(d_spikes_travelling_to_synapse,current_time_in_seconds,circular_spikenum_buffer,spikeid_buffer,bufferloc,buffersize,total_number_of_synapses,d_time_of_last_spike_to_reach_synapse,postsynaptic_neuron_indices,neuron_wise_conductance_trace,synaptic_decay_id,total_number_of_neurons,d_synaptic_efficacies_or_weights,d_biological_conductance_scaling_constants_lambda,timestep);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}