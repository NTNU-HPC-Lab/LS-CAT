#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assemble_boundary_potential_on_device(float * d_potential_dot_dot_acoustic, const float * d_send_potential_dot_dot_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
int id;
int iglob;
int iloc;
int iinterface;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + ((gridDim.x) * (blockDim.x)) * (threadIdx.y + (blockIdx.y) * (blockDim.y));
for (iinterface = 0; iinterface <= num_interfaces - (1); iinterface += 1) {
if (id < d_nibool_interfaces[iinterface]) {
iloc = id + (max_nibool_interfaces) * (iinterface);
iglob = d_ibool_interfaces[iloc] - (1);
atomicAdd(d_potential_dot_dot_acoustic + iglob, d_send_potential_dot_dot_buffer[iloc]);
}
}
}