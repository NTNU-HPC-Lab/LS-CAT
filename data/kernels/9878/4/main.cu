#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "assemble_boundary_potential_on_device.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_potential_dot_dot_acoustic = NULL;
hipMalloc(&d_potential_dot_dot_acoustic, XSIZE*YSIZE);
const float *d_send_potential_dot_dot_buffer = NULL;
hipMalloc(&d_send_potential_dot_dot_buffer, XSIZE*YSIZE);
const int num_interfaces = 1;
const int max_nibool_interfaces = 1;
const int *d_nibool_interfaces = NULL;
hipMalloc(&d_nibool_interfaces, XSIZE*YSIZE);
const int *d_ibool_interfaces = NULL;
hipMalloc(&d_ibool_interfaces, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
assemble_boundary_potential_on_device<<<gridBlock,threadBlock>>>(d_potential_dot_dot_acoustic,d_send_potential_dot_dot_buffer,num_interfaces,max_nibool_interfaces,d_nibool_interfaces,d_ibool_interfaces);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
assemble_boundary_potential_on_device<<<gridBlock,threadBlock>>>(d_potential_dot_dot_acoustic,d_send_potential_dot_dot_buffer,num_interfaces,max_nibool_interfaces,d_nibool_interfaces,d_ibool_interfaces);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
assemble_boundary_potential_on_device<<<gridBlock,threadBlock>>>(d_potential_dot_dot_acoustic,d_send_potential_dot_dot_buffer,num_interfaces,max_nibool_interfaces,d_nibool_interfaces,d_ibool_interfaces);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}