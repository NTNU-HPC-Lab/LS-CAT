#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepare_boundary_potential_on_device(const float * d_potential_dot_dot_acoustic, float * d_send_potential_dot_dot_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
int id;
int iglob;
int iloc;
int iinterface;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + ((gridDim.x) * (blockDim.x)) * (threadIdx.y + (blockIdx.y) * (blockDim.y));
for (iinterface = 0; iinterface <= num_interfaces - (1); iinterface += 1) {
if (id < d_nibool_interfaces[iinterface]) {
iloc = id + (max_nibool_interfaces) * (iinterface);
iglob = d_ibool_interfaces[iloc] - (1);
d_send_potential_dot_dot_buffer[iloc] = d_potential_dot_dot_acoustic[iglob];
}
}
}