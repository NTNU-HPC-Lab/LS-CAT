#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_accel_elastic_kernel(float * accel, const float * veloc, const int size, const float two_omega_earth, const float * rmassx, const float * rmassy, const float * rmassz){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
accel[(id) * (3)] = (accel[(id) * (3)]) * (rmassx[id]) + (two_omega_earth) * (veloc[(id) * (3) + 1]);
accel[(id) * (3) + 1] = (accel[(id) * (3) + 1]) * (rmassy[id]) - ((two_omega_earth) * (veloc[(id) * (3)]));
accel[(id) * (3) + 2] = (accel[(id) * (3) + 2]) * (rmassz[id]);
}
}