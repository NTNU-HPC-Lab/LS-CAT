#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_veloc_acoustic_kernel(float * veloc, const float * accel, const int size, const float deltatover2){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
veloc[id] = veloc[id] + (deltatover2) * (accel[id]);
}
}