#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_veloc_elastic_kernel(float * veloc, const float * accel, const int size, const float deltatover2){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
veloc[id] = veloc[id] + (deltatover2) * (accel[id]);
veloc[size + id] = veloc[size + id] + (deltatover2) * (accel[size + id]);
veloc[size + size + id] = veloc[size + size + id] + (deltatover2) * (accel[size + size + id]);
}
}