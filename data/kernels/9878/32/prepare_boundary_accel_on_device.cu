#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepare_boundary_accel_on_device(const float * d_accel, float * d_send_accel_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
int id;
int iglob;
int iloc;
int iinterface;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + ((gridDim.x) * (blockDim.x)) * (threadIdx.y + (blockIdx.y) * (blockDim.y));
for (iinterface = 0; iinterface <= num_interfaces - (1); iinterface += 1) {
if (id < d_nibool_interfaces[iinterface]) {
iloc = id + (max_nibool_interfaces) * (iinterface);
iglob = d_ibool_interfaces[iloc] - (1);
d_send_accel_buffer[(iloc) * (3) + 0] = d_accel[(iglob) * (3) + 0];
d_send_accel_buffer[(iloc) * (3) + 1] = d_accel[(iglob) * (3) + 1];
d_send_accel_buffer[(iloc) * (3) + 2] = d_accel[(iglob) * (3) + 2];
}
}
}