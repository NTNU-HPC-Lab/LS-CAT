#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_accel_acoustic_kernel(float * accel, const int size, const float * rmass){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
accel[id] = (accel[id]) * (rmass[id]);
}
}