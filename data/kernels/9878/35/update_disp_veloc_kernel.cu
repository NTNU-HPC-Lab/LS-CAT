#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_disp_veloc_kernel(float * displ, float * veloc, float * accel, const int size, const float deltat, const float deltatsqover2, const float deltatover2){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
displ[id] = displ[id] + (deltat) * (veloc[id]) + (deltatsqover2) * (accel[id]);
veloc[id] = veloc[id] + (deltatover2) * (accel[id]);
accel[id] = 0.0f;
}
}