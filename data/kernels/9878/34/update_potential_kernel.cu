#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_potential_kernel(float * potential_acoustic, float * potential_dot_acoustic, float * potential_dot_dot_acoustic, const int size, const float deltat, const float deltatsqover2, const float deltatover2){
int id;
id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
if (id < size) {
potential_acoustic[id] = potential_acoustic[id] + (deltat) * (potential_dot_acoustic[id]) + (deltatsqover2) * (potential_dot_dot_acoustic[id]);
potential_dot_acoustic[id] = potential_dot_acoustic[id] + (deltatover2) * (potential_dot_dot_acoustic[id]);
potential_dot_dot_acoustic[id] = 0.0f;
}
}