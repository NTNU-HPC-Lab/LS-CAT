#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength){
float	r_u1;
float	r_v1;
float	r_d1,r_dim=0;

r_dim=0;
for(int d=0; d<pointdim; d++){
r_u1 = *(g_uquery+d*signallength);
r_v1 = *(g_vpoint+d*signallength);
r_d1 = r_v1 - r_u1;
r_d1 = r_d1 < 0? -r_d1: r_d1;  //abs
r_dim= r_dim < r_d1? r_d1: r_dim;
}
return r_dim;
}
__device__ float insertPointKlist(int kth, float distance, int indexv,float* kdistances, int* kindexes){
int k=0;
while( (distance>*(kdistances+k)) && (k<kth-1)){k++;}
//Move value to the next
for(int k2=kth-1;k2>k;k2--){
*(kdistances+k2)=*(kdistances+k2-1);
*(kindexes+k2)=*(kindexes+k2-1);
}
//Replace
*(kdistances+k)=distance;
*(kindexes+k)=indexv;

//printf("\n -> Modificacion pila: %.f %.f. New max distance: %.f", *kdistances, *(kdistances+1), *(kdistances+kth-1));
return *(kdistances+kth-1);
}
__global__ void kernelKNN(const float* g_uquery, const float* g_vpointset, int *g_indexes, float* g_distances, int pointdim, int triallength, int signallength, int kth, int exclude)
{

const unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
//const unsigned int tidim = tid*pointdim;
const unsigned int itrial = tid / triallength;  //  indextrial

int kindexes[] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
float kdistances[]= {INFINITY, INFINITY, INFINITY, INFINITY, INFINITY, \
INFINITY, INFINITY, INFINITY, INFINITY, INFINITY};

if(tid<signallength){

//int   r_index;
float r_kdist=INFINITY;
int indexi = tid-triallength*itrial;
for(int t=0; t<triallength; t++){
int indexu = tid;
int indexv = (t + itrial*triallength);
int condition1=indexi-exclude;
int condition2=indexi+exclude;
if((t<condition1)||(t>condition2)){
float temp_dist = maxMetricPoints(g_uquery+indexu, g_vpointset+indexv,pointdim, signallength);
if(temp_dist <= r_kdist){
r_kdist = insertPointKlist(kth,temp_dist,t,kdistances,kindexes);
//printf("\nId: %d, Temp_dist: %.f. r_index: %d", tid, temp_dist, r_index);
}
}
//printf("tid:%d indexes: %d, %d distances: %.f %.f\n",tid, *kindexes, *(kindexes+1), *kdistances, *(kdistances+1));
}

__syncthreads();
//COPY TO GLOBAL MEMORY
for(int k=0;k<kth;k++){
g_indexes[tid+k*signallength] = *(kindexes+k);
g_distances[tid+k*signallength]= *(kdistances+k);
}

}

}