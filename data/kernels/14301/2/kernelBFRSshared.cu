#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength){
float	r_u1;
float	r_v1;
float	r_d1,r_dim=0;

r_dim=0;
for(int d=0; d<pointdim; d++){
r_u1 = *(g_uquery+d*signallength);
r_v1 = *(g_vpoint+d*signallength);
r_d1 = r_v1 - r_u1;
r_d1 = r_d1 < 0? -r_d1: r_d1;  //abs
r_dim= r_dim < r_d1? r_d1: r_dim;
}
return r_dim;
}
__global__ void kernelBFRSshared(const float* g_uquery, const float* g_vpointset, int *g_npoints, int pointdim, int triallength, int signallength, int exclude, float radius)
{

// shared memory
extern __shared__ char array[];
int *s_npointsrange;
s_npointsrange = (int*)array;

const unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
const unsigned int itrial = tid / triallength;  //  indextrial

if(tid<signallength){

s_npointsrange[threadIdx.x] = 0;
__syncthreads();


unsigned int indexi = tid-triallength*itrial;
for(int t=0; t<triallength; t++){
int indexu = tid;
int indexv = (t + itrial*triallength);
int condition1=indexi-exclude;
int condition2=indexi+exclude;
if((t<condition1)||(t>condition2)){
float temp_dist = maxMetricPoints(g_uquery+indexu, g_vpointset+indexv,pointdim, signallength);
if(temp_dist <= radius){
s_npointsrange[threadIdx.x]++;
}
}

}

__syncthreads();
//printf("\ntid:%d npoints: %d\n",tid, s_npointsrange[threadIdx.x]);
//COPY TO GLOBAL MEMORY
g_npoints[tid] = s_npointsrange[threadIdx.x];

}
}