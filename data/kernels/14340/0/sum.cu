#include "hip/hip_runtime.h"
#include "includes.h"
/**
* This is an example from the paper "Formal Semantics of Heterogeneous CUDA-C:
* A Modular Approach with Applications" by Chris Hathhorn et al.
*/


#define N 8
#define NBLOCKS 4
#define NTHREADS (N/NBLOCKS)


__global__ void sum(int* in, int* out) {
extern __shared__ int shared[];
int i, tid = threadIdx.x,
bid = blockIdx.x,
bdim = blockDim.x;

shared[tid] = in[bid * bdim + tid];

__syncthreads();
if(tid < bdim/2) {
shared[tid] += shared[bdim/2 + tid];
}
__syncthreads();
if(tid == 0) {
for (i = 1; i != (bdim/2) + (bdim%2); ++i) {
shared[0] += shared[i];
}
out[bid] = shared[0];
}
}