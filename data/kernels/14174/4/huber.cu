#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void huber(float *a, const size_t width, const size_t height, const float alpha, const float strength, const size_t pixelsPerThread, float *f)
{
const size_t col  = (blockIdx.x * blockDim.x + threadIdx.x) % width;
const size_t crow = (blockIdx.x * blockDim.x + threadIdx.x) / width * pixelsPerThread;

if (col >= width || crow >= height)
return;

const size_t erow = min((unsigned int)(crow + pixelsPerThread), (unsigned int)height);

const float alpha2 = alpha * alpha;

float colF = 0.0f;

for (size_t row = crow; row < erow; ++row)
{
const size_t idx = row * width + col;

// Pseudo-Huber loss function
const float root = sqrtf(1.0f + a[idx]*a[idx] / alpha2);
colF += alpha2 * (root - 1.0f);
a[idx] *= strength / root;
}

colF *= strength;
f[blockIdx.x * blockDim.x + threadIdx.x] = colF;
}