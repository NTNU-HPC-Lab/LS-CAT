#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divideByCSCColSums(const float *values, const int *colPointers, float *pixels, const size_t n)
{
const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= n)
return;

float weight = 0.0f;

for (size_t ridx = colPointers[idx]; ridx < colPointers[idx+1]; ++ridx)
{
weight += values[ridx];
}

pixels[idx] /= weight + 1e-6f;
}