#include "hip/hip_runtime.h"
#include "includes.h"
__device__ static void myAtomicAdd(float *address, float value)
{
#if __CUDA_ARCH__ >= 200
atomicAdd(address, value);
#else
// cf. https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
int oldval, newval, readback;

oldval = __float_as_int(*address);
newval = __float_as_int(__int_as_float(oldval) + value);
while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval)
{
oldval = readback;
newval = __float_as_int(__int_as_float(oldval) + value);
}
#endif
}
__global__ void computeCSRColSums(float *d_colSums, const float *d_systemMatrixVals, const int *d_systemMatrixRows, const int *d_systemMatrixCols, const size_t m, const size_t n)
{
const size_t row = blockIdx.x * blockDim.x + threadIdx.x;

if (row >= m)
return;

for (size_t cidx = d_systemMatrixRows[row]; cidx < d_systemMatrixRows[row+1]; ++cidx)
{
myAtomicAdd(d_colSums + d_systemMatrixCols[cidx], d_systemMatrixVals[cidx]);
}
}