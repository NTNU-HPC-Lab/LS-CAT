#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int roundToInt(float val)
{
return (int)floor(val + 0.5f);
}
__device__ float d_priorF;  __global__ void add(float *p, float *q) { *p += *q; }
__global__ void composeSingleSystem(const size_t offset, const float *H, const size_t lowresWidth,  const size_t lowresHeight, const size_t highresWidth, const size_t highresHeight, const float psfWidth, const int pixelRadius, float *systemMatrixVals, int *systemMatrixCols, int *systemMatrixRows)
{
const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

const size_t lowresPixels  = lowresWidth  * lowresHeight;

if (idx >= lowresPixels)
return;

// Coordinates of this thread in the low-res image
size_t x = idx % lowresWidth;
size_t y = idx / lowresWidth;

// Row that this thread writes in the full system matrix
size_t r = idx + offset;

// Transform pixel coordinates from the LR grid to the desired HR grid

float hrx, hry;
float zoom = float(highresWidth) / float(lowresWidth);

hrx = (H[0] * x + H[1] * y + H[2]) * zoom;
hry = (H[3] * x + H[4] * y + H[5]) * zoom;

float weightSum = 0.0f;

const size_t maxRowElems = (2 * pixelRadius + 1) * (2 * pixelRadius + 1);
size_t offsetCRS = 0;
size_t offsetRows = maxRowElems * r;

// Iterate over the neighborhood defined by the width of the psf
for (int offsetY = -pixelRadius; offsetY <= pixelRadius; ++offsetY)
{
const int ny = roundToInt(hry + offsetY);

if (ny < 0 || ny >= highresHeight)
continue;

for (int offsetX = -pixelRadius; offsetX <= pixelRadius; ++offsetX)
{
const int nx = roundToInt(hrx + offsetX);

if (nx < 0 || nx >= highresWidth)
continue;

const float dx = hrx - float(nx);
const float dy = hry - float(ny);

// Compute influence of current high-res pixel for
// this thread's low-res pixel

float dist = dx*dx*H[0]*H[0] + dy*dy*H[4]*H[4] +
dx*dy*H[0]*H[3] + dx*dy*H[1]*H[4];

float weight = expf(-dist / (2.0f * zoom * zoom * psfWidth * psfWidth));

const size_t valIdx = offsetRows + offsetCRS;
systemMatrixVals[valIdx] = weight;
systemMatrixCols[valIdx] = ny * highresWidth + nx;

weightSum += weight;

++offsetCRS;
}
}

if (weightSum > 0.0f)
{
// Normalize row sums
for (size_t i = 0; i < offsetCRS; ++i)
{
systemMatrixVals[offsetRows + i] /= weightSum;
}
}

// If we have saved less than maxRowElems elements,
// we have to pad the CRS structure with 0 entries
// to make sure it is valid

if (offsetCRS == 0)
{
systemMatrixVals[offsetRows] = 0.0f;
systemMatrixCols[offsetRows] = 0;
++offsetCRS;
}

bool copy = false;

// Try adding elements after the last saved entry

while (offsetCRS < maxRowElems)
{
const size_t idx = offsetRows + offsetCRS;

if (systemMatrixCols[idx - 1] + 1 >= highresWidth * highresHeight)
{
copy = true;
break;
}

systemMatrixVals[idx] = 0.0f;
systemMatrixCols[idx] = systemMatrixCols[idx - 1] + 1;
offsetCRS++;
}

// If there isn't enough space after the last saved
// entry, add padding before first entry

if (copy)
{
for (int idx = offsetCRS - 1; idx >= 0; --idx)
{
systemMatrixVals[offsetRows + maxRowElems - (offsetCRS - idx)] =
systemMatrixVals[offsetRows + idx];
systemMatrixCols[offsetRows + maxRowElems - (offsetCRS - idx)] =
systemMatrixCols[offsetRows + idx];
}

for (int idx = maxRowElems - offsetCRS - 1; idx >= 0; --idx)
{
systemMatrixVals[offsetRows + idx] = 0.0f;
systemMatrixCols[offsetRows + idx] = systemMatrixCols[offsetRows + idx + 1] - 1;
}
}

systemMatrixRows[r] = r * maxRowElems;
}