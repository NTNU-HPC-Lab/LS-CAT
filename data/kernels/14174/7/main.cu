#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "composeSingleSystem.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const size_t offset = XSIZE*YSIZE;
const float *H = NULL;
hipMalloc(&H, XSIZE*YSIZE);
const size_t lowresWidth = XSIZE*YSIZE;
const size_t lowresHeight = XSIZE*YSIZE;
const size_t highresWidth = XSIZE*YSIZE;
const size_t highresHeight = XSIZE*YSIZE;
const float psfWidth = 1;
const int pixelRadius = 1;
float *systemMatrixVals = NULL;
hipMalloc(&systemMatrixVals, XSIZE*YSIZE);
int *systemMatrixCols = NULL;
hipMalloc(&systemMatrixCols, XSIZE*YSIZE);
int *systemMatrixRows = NULL;
hipMalloc(&systemMatrixRows, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
composeSingleSystem<<<gridBlock,threadBlock>>>(offset,H,lowresWidth,lowresHeight,highresWidth,highresHeight,psfWidth,pixelRadius,systemMatrixVals,systemMatrixCols,systemMatrixRows);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
composeSingleSystem<<<gridBlock,threadBlock>>>(offset,H,lowresWidth,lowresHeight,highresWidth,highresHeight,psfWidth,pixelRadius,systemMatrixVals,systemMatrixCols,systemMatrixRows);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
composeSingleSystem<<<gridBlock,threadBlock>>>(offset,H,lowresWidth,lowresHeight,highresWidth,highresHeight,psfWidth,pixelRadius,systemMatrixVals,systemMatrixCols,systemMatrixRows);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}