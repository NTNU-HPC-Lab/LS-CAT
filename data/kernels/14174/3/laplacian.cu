#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void laplacian(float *dst, const float *src, const size_t width, const size_t height, const size_t pixelsPerThread)
{
const size_t col  = (blockIdx.x * blockDim.x + threadIdx.x) % width;
const size_t crow = (blockIdx.x * blockDim.x + threadIdx.x) / width * pixelsPerThread;

if (col >= width || crow >= height)
return;

const size_t srow = crow + 1;
const size_t erow = min((unsigned int)(crow + pixelsPerThread - 1), (unsigned int)(height - 1));

// First element

const size_t firstIdx = crow * width + col;

dst[firstIdx] = src[firstIdx];

if (crow + 1 <  height) dst[firstIdx] -= 0.25f * src[firstIdx + width]; // S
if (crow     >= 1)      dst[firstIdx] -= 0.25f * src[firstIdx - width]; // N
if (col + 1  <  width)  dst[firstIdx] -= 0.25f * src[firstIdx + 1]; // E
if (col      >= 1)      dst[firstIdx] -= 0.25f * src[firstIdx - 1]; // W

// Inner elements

for (int row = srow; row < erow; ++row)
{
const size_t cIdx = row * width + col;

// C, S, N (always exist)
dst[cIdx] = src[cIdx] - 0.25f * (src[cIdx + width] + src[cIdx - width]);

if (col + 1 < width) dst[cIdx] -= 0.25f * src[cIdx + 1]; // E
if (col     >= 1)    dst[cIdx] -= 0.25f * src[cIdx - 1]; // W
}

if (erow <= crow)
return;

// Last element

const size_t lastIdx = erow * width + col;

dst[lastIdx] = src[lastIdx] - 0.25f * src[lastIdx - width]; // C, N

if (erow + 1 <  height) dst[lastIdx] -= 0.25f * src[lastIdx + width]; // S
if (col + 1  <  width)  dst[lastIdx] -= 0.25f * src[lastIdx + 1]; // E
if (col      >= 1)      dst[lastIdx] -= 0.25f * src[lastIdx - 1]; // W
}