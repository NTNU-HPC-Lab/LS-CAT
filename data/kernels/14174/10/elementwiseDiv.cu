#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwiseDiv(float *a, const float *b, const size_t len)
{
const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= len)
return;

a[idx] /= b[idx] + 1e-6f;
}