#include "hip/hip_runtime.h"
#include "includes.h"



#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)




__global__ void erodeDepthMapDevice(float* d_output, float* d_input, int structureSize, int width, int height, float dThresh, float fracReq)
{
const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;


if (x >= 0 && x < width && y >= 0 && y < height)
{


unsigned int count = 0;

float oldDepth = d_input[y*width + x];
for (int i = -structureSize; i <= structureSize; i++)
{
for (int j = -structureSize; j <= structureSize; j++)
{
if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
{
float depth = d_input[(y + i)*width + (x + j)];
if (depth == MINF || depth == 0.0f || fabs(depth - oldDepth) > dThresh)
{
count++;
//d_output[y*width+x] = MINF;
//return;
}
}
}
}

unsigned int sum = (2 * structureSize + 1)*(2 * structureSize + 1);
if ((float)count / (float)sum >= fracReq) {
d_output[y*width + x] = MINF;
}
else {
d_output[y*width + x] = d_input[y*width + x];
}
}
}