#include "hip/hip_runtime.h"
#include "includes.h"



#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)




__global__ void resampleUCHAR4_Kernel(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x < outputWidth && y < outputHeight)
{
const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

if (xInput < inputWidth && yInput < inputHeight) {
d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
}
}
}