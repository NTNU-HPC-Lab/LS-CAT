#include "hip/hip_runtime.h"
#include "includes.h"



#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)




__global__ void computeIntensityDerivatives_Kernel(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x < width && y < height)
{
d_output[y*width + x] = make_float2(MINF, MINF);

//derivative
if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
{
float pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00 == MINF) return;
float pos01 = d_input[(y - 0)*width + (x - 1)];	if (pos01 == MINF) return;
float pos02 = d_input[(y + 1)*width + (x - 1)];	if (pos02 == MINF) return;

float pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10 == MINF) return;
//float pos11 = d_input[(y-0)*width + (x-0)]; if (pos11 == MINF) return;
float pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12 == MINF) return;

float pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20 == MINF) return;
float pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21 == MINF) return;
float pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22 == MINF) return;

float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
(-2.0f)*pos01 + (2.0f)*pos21 +
(-1.0f)*pos02 + (1.0f)*pos22;
resU /= 8.0f;

float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
resV /= 8.0f;

d_output[y*width + x] = make_float2(resU, resV);
}
}
}