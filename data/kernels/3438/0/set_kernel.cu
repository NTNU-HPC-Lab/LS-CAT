#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_kernel(int* dst, int const value, int const count)
{
int const index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= count)
return;

dst[index] = value;
}