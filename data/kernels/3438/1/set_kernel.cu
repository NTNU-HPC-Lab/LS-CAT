#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_kernel(REAL* dst, REAL const value, std::size_t const count)
{
std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

if (index >= count)
return;

dst[index] = value;
}