#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/



#define SIZE    (100*1024*1024)



__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {

// clear out the accumulation buffer called temp
// since we are launched with 256 threads, it is easy
// to clear that memory with one write per thread
__shared__  unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

// calculate the starting index and the offset to the next
// block that each thread will be processing
int i = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while (i < size) {
atomicAdd( &temp[buffer[i]], 1 );
i += stride;
}
// sync the data from the above writes to shared memory
// then add the shared memory values to the values from
// the other thread blocks using global memory
// atomic adds
// same as before, since we have 256 threads, updating the
// global histogram is just one write per thread!
__syncthreads();
atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}