#include "hip/hip_runtime.h"
#include "includes.h"
/*Title: Vector addition and subtraction in CUDA.
A simple way to understand how CUDA can be used to perform arithmetic operations.
*/
using namespace std;
# define size 5

//Global functions

//********************************************************
__global__ void AddIntsCUDA(int *a, int *b)
{
int tid=blockIdx.x*blockDim.x+threadIdx.x;
a[tid] = a[tid] + b[tid];
}