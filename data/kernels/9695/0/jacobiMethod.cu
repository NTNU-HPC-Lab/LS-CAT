#include "hip/hip_runtime.h"
#include "includes.h"
//============================================================================
// Name        : PoissonEquationJacobiCuda.cpp
// Author      :
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================


using namespace std;

const float PI = 4*atan(1);

__global__ void jacobiMethod(float* grid,float* potential, int sizeX,int sizeY,float scale,int noIters,float tolerance){

extern __shared__ float sharedMem[];
/*
Shared memory
1st part is grid
2nd part is initial guess
3rd part is current Solution

*/


// Copying from global to shared memory
int threadIdX = threadIdx.x;
int threadIdY = threadIdx.y;

if (threadIdX == 0 && threadIdY == 0) {
//printf("At Beginning\n");
}


int bOx = blockIdx.x * blockDim.x;
int bOy = blockIdx.y * blockDim.y;

//int totalBlockThreadId = threadIdY*blockDim.x + threadIdX;

//int blockThreadIdx = threadIdX-noIters;
//int blockThreadIdy = threadIdY-noIters;

int effBlockSizeX = blockDim.x + 2 * noIters;
int effBlockSizeY = blockDim.y + 2 * noIters;


int totalSize = sizeX*sizeY;

int sharedMemSize = effBlockSizeX*effBlockSizeY;

for(int i= threadIdX;i<effBlockSizeX;i+= blockDim.x)
for (int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
int currElemSM = i*effBlockSizeX + j;
int currElemMain = (i - noIters + bOy)*sizeX + (j - noIters + bOx);
if (currElemMain >= 0 && currElemMain < totalSize) {
sharedMem[currElemSM] = grid[currElemMain];
sharedMem[currElemSM + sharedMemSize] = potential[currElemMain];
}
else {
sharedMem[currElemSM] = 0;
sharedMem[currElemSM + sharedMemSize] = 0;
}
sharedMem[currElemSM + 2 * sharedMemSize] = 0;
}
__syncthreads();
if (threadIdX == 0 && threadIdY == 0) {
//printf("Copied to shared memory\n");
}

for(int k=0;k<noIters;k++){
for(int i= threadIdX;i<effBlockSizeX;i+= blockDim.x)
for(int j= threadIdY;j<effBlockSizeY;j+= blockDim.y){
int currPos = i*effBlockSizeX +j+ sharedMemSize*2;
sharedMem[currPos]=0;
if(i>1){
sharedMem[currPos]+=(sharedMem[currPos- effBlockSizeY- sharedMemSize]/4);
}
if(i<effBlockSizeX -1){
sharedMem[currPos]+=(sharedMem[currPos+ effBlockSizeY - sharedMemSize]/4);
}
if(j>1){
sharedMem[currPos]+=(sharedMem[currPos-1- sharedMemSize]/4);
}
if(j<effBlockSizeY-1){
sharedMem[currPos]+=(sharedMem[currPos+1- sharedMemSize]/4);
}
if(i== effBlockSizeX-1||j== effBlockSizeY-1){
//currSolution[currPos]=0;
}else if(currPos - 2 * sharedMemSize>=0){
sharedMem[currPos]+=(scale*scale/4* sharedMem[currPos-2* sharedMemSize]);
}
}
__syncthreads();

for (int i = threadIdX; i<effBlockSizeX; i += blockDim.x)
for (int j = threadIdY; j<effBlockSizeY; j += blockDim.y) {
int currPos = i*effBlockSizeX + j + sharedMemSize * 2;
sharedMem[currPos- sharedMemSize]= sharedMem[currPos];
}
__syncthreads();
}
if (threadIdX == 0 && threadIdY == 0) {
//printf("Done computation\n");
}

for (int i = threadIdX; i<effBlockSizeX; i += blockDim.x)
for (int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
if (i >= noIters && j >= noIters && i < effBlockSizeX - noIters && j < effBlockSizeX - noIters) {
int currElemSM = i*effBlockSizeX + j;
int currElemMain = (i - noIters + bOy)*sizeX + (j - noIters + bOx);
if (currElemMain > 0 && currElemMain < totalSize) {
potential[currElemMain] = sharedMem[currElemSM + 2 * sharedMemSize];
}

}
}
if (threadIdX == 0 && threadIdY == 0) {
//printf("Copied to memory\n");
}

}