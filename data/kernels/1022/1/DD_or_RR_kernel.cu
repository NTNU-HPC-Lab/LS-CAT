#include "hip/hip_runtime.h"
#include "includes.h"


// 2-point angular correlation

const int BLOCKSIZE = 256;
const int ROWSPERTHREAD = 256;

// Columns are D and rows are R

// All computation in single-precision

__global__ void DD_or_RR_kernel(int nCols, int nRows, float *arr, unsigned long long int *gHist) {

// The thread id on the x-axis and y-axis
//int x = blockIdx.y * ROWSPERTHREAD + blockIdx.x * blockDim.x + threadIdx.x;
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * ROWSPERTHREAD;

// If the column is inside the domain and the last row of the thread should be computed
if (x < nCols && y + ROWSPERTHREAD > x) {

// Shared histogram for the thread block
__shared__ unsigned int sHist[720];

// Thread number zero will initialize the shared memory
if (threadIdx.x == 0) {
for (int i = 0; i < 720; i++) {
sHist[i] = 0;
}
}

__syncthreads();

// Right ascension and declination in degrees for the current column
float asc1 = arr[x * 2];
float dec1 = arr[x * 2 + 1];

// Offset is at which row to start computing
int offset = max(x-y+1, 0);

// The amount of rows to be calculated is ROWSPERTHREAD or rows left in the domain, whichever is smaller
int nElements = min(nRows-y, ROWSPERTHREAD);

for (int j = offset; j < nElements; j++) {
// Right ascension and declination in degrees for the current row
float asc2 = arr[(y + j) * 2];
float dec2 = arr[(y + j) * 2 + 1];

// Compute the intermediate value
float tmp = sinf(dec1) * sinf(dec2) + cosf(dec1) * cosf(dec2) * cosf(asc1-asc2);

// Clamp it to -1, 1
tmp = fminf(tmp, 1.0f);
tmp = fmaxf(tmp, -1.0f);

// Compute the angle in radians
float radianResult = acosf(tmp);

// Convert to degrees
float degreeResult = radianResult * 180.0f/3.14159f;

// Compute the bin index
int resultIndex = floor(degreeResult * 4.0f);

// Increment the bin in the shared histogram
atomicAdd(&sHist[resultIndex], 2);
}

__syncthreads();

// Thread number zero will write the shared histogram to global device memory
if (threadIdx.x == 0) {
for (int i = 0; i < 720; i++) {
// Update the global histogram with the shared histogram
atomicAdd(&gHist[i], sHist[i]);
}
}
}
}