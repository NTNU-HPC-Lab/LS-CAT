#include "hip/hip_runtime.h"
#include "includes.h"


// 2-point angular correlation

const int BLOCKSIZE = 256;
const int ROWSPERTHREAD = 256;

// Columns are D and rows are R

// All computation in single-precision

__global__ void DR_kernel(int nCols, int nRows, float *D, float *R, unsigned long long int *gHist) {

// The thread id on the x-axis and y-axis
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * ROWSPERTHREAD;

// If the thread is inside the domain
if (x < nCols) {

// Shared histogram for the thread block
__shared__ unsigned int sHist[720];

// Thread number zero will initialize the shared memory
if (threadIdx.x == 0) {
for (int i = 0; i < 720; i++) {
sHist[i] = 0;
}
}

__syncthreads();

// Right ascension and declination in degrees for the current column
float asc1 = D[x * 2];
float dec1 = D[x * 2 + 1];

// The amount of rows to be calculated is ROWSPERTHREAD or rows left in the domain, whichever is smaller
int nElements = min(nRows-y, ROWSPERTHREAD);

for (int j = 0; j < nElements; j++) {
// Right ascension and declination degrees for the current row
float asc2 = R[y + j * 2];
float dec2 = R[y + j * 2 + 1];

// Compute the intermediate value
float tmp = sinf(dec1) * sinf(dec2) + cosf(dec1) * cosf(dec2) * cosf(asc1-asc2);

// Clamp it to -1, 1
tmp = fminf(tmp, 1.0f);
tmp = fmaxf(tmp, -1.0f);

// Compute the angle in radians
float radianResult = acosf(tmp);

// Convert to degrees
float degreeResult = radianResult * 180.0f/3.14159f;

// Compute the bin index
int resultIndex = floor(degreeResult * 4.0f);

// Increment the bin in the shared histogram
atomicAdd(&sHist[resultIndex], 1);

}

__syncthreads();

// Thread number zero will write the shared histogram to global device memory
if (threadIdx.x == 0) {
for (int i = 0; i < 720; i++) {
// Update the global histogram with the shared histogram
atomicAdd(&gHist[i], sHist[i]);
}
}
}
}