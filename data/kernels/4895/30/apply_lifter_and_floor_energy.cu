#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
int thread_id = threadIdx.x;
int frame = blockIdx.x;

float *feats = features + frame * ldf;

// apply lifter coefficients
if (cepstral_lifter != 0.0f) {
for (int c = thread_id; c < num_cols; c += CU1DBLOCK) {
float lift = lifter_coeffs[c];
float f = feats[c];
feats[c] = f * lift;
}
}

// Thread 0 for each frame will apply energy
if (use_energy && thread_id == 0) {
float energy = log_energy[frame];
float log_energy_floor = log(energy_floor);

if (energy_floor > 0.0f && energy < log_energy_floor) {
energy = log_energy_floor;
}
feats[0] = energy;
}
}