#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel( float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride, int paddingValue) {

int idx = blockIdx.x * 4 + threadIdx.y;

// Each warp is responsible for an input into the LookupTable.
// If the preceeding input has the same as this input, then the warp
// exits immediately. The warp also processes subsequent inputs with the
// same value.
//
// Input Warp
// 1     <warp 1>
// 1     <warp 1> (<warp 2> exits without doing any work)
// 5     <warp 3>
// 8     <warp 4>

// Number of values proceessed by each thread (grain size)
const int SZ = 4;

if (idx < numel
&& (idx == 0 || input[idx] != input[idx - 1])
&& input[idx] != paddingValue) {
do {
const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
const int weightRow = ((int) input[idx] - 1) * stride;
const int gradOutputRow = ((int) indices[idx] - 1) * stride;
const float scale = count ? defaultScale / count[idx] : defaultScale;

float gradient[SZ];
float weight[SZ];

#pragma unroll
for (int ii = 0; ii < SZ; ii++)
{
int featureDim = startFeature + ii * WARP_SIZE;
if (featureDim < stride)
{
gradient[ii] = gradOutput[gradOutputRow + featureDim];
weight[ii] = gradWeight[weightRow + featureDim];
}
}

#pragma unroll
for (int ii = 0; ii < SZ; ii++)
{
weight[ii] += gradient[ii] * scale;
}

#pragma unroll
for (int ii = 0; ii < SZ; ii++)
{
int featureDim = startFeature + ii * WARP_SIZE;
if (featureDim < stride)
{
gradWeight[weightRow + featureDim] = weight[ii];
}
}

idx++;
} while (idx < numel && input[idx] == input[idx - 1]);
}
}