#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float *a, float *b, float *c)
{
__shared__ float cache[threadsPerBlock];
int cacheIndex = threadIdx.x;

float temp = 0.0;
for (int tid = threadIdx.x + blockIdx.x*blockDim.x; tid<N; tid += blockDim.x*gridDim.x)
{
temp += a[tid]*b[tid];
}

cache[cacheIndex] = temp;

__syncthreads();


// reduction
for (int i = blockDim.x/2; i>0; i /= 2)
{
if (cacheIndex < i) cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
}

if (threadIdx.x == 0) c[blockIdx.x] = cache[0];
}