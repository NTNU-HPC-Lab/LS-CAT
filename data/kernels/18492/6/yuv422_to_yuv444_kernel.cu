#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void yuv422_to_yuv444_kernel(const void * src, void * out, int pix_count) {
// coordinates of this thread
const int block_idx_x = threadIdx.x + blockIdx.x * blockDim.x;

// skip if out of bounds
if(block_idx_x >= pix_count / 2) {
return;
}

uchar4 *this_src = ((uchar4 *) src) + block_idx_x * 2;
uchar4 *this_out = ((uchar4 *) out) + block_idx_x * 3;

uchar4 pix12 = this_src[0];
uchar4 pix34 = this_src[1];

uchar4 out_pix[3];
out_pix[0].x = pix12.y;
out_pix[0].y = pix12.x;
out_pix[0].z = pix12.z;

out_pix[0].w = pix12.w;
out_pix[1].x = pix12.x;
out_pix[1].y = pix12.z;

out_pix[1].z = pix34.y;
out_pix[1].w = pix34.x;
out_pix[2].x = pix34.z;

out_pix[2].y = pix34.w;
out_pix[2].z = pix34.x;
out_pix[2].w = pix34.z;

this_out[0] = out_pix[0];
this_out[1] = out_pix[1];
this_out[2] = out_pix[2];
}