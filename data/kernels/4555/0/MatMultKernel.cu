#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 1
//Global variable set up
const int radius = 3;
const int numSamples = 100;
const double learningRate = 0.15;
const int epochs = 1;
const int numNeurons =20;
//Set up neurons
double inputLayer[2][numNeurons] = { 0 }; //takes input and weights
double outputLayer[1][numNeurons] = { 0 }; //takes weights and outputs
double * matrixA; //Temporarily initialised arrays which are allocated to aid in gpu memory allocation
double * matrixB;
double * matrixC;
double * matrixD;
double * matrixE;

//Calculates dot product of two arrays from a given pointer and returns a total - must be same size
__global__ void MatMultKernel(double *array1, double *array2, double *output, int arr1_rows, int arr1_cols, int arr2_cols) {
double result = 0;
__shared__ double subArray1[BLOCK_SIZE][BLOCK_SIZE]; //Setting up the shared memory into sub arrays for more efficient computation
__shared__ double subArray2[BLOCK_SIZE][BLOCK_SIZE];
int bIDx = blockIdx.x, bIDy = blockIdx.y, tIDx = threadIdx.x, tIDy = threadIdx.y; //Setting up variables to identify threads uniquely
int row = bIDy * BLOCK_SIZE + tIDy; //Setting the given row of a thread
int col = bIDx * BLOCK_SIZE + tIDx; //Setting the given col of a thread
for (int i = 0; i < (arr1_cols-1)/BLOCK_SIZE+1; i++) { //Iterating through every chunk of columns proportional to block size
if (row < arr1_rows && i*BLOCK_SIZE+tIDx<arr1_cols) {
subArray1[tIDy][tIDx] = array1[row*arr1_cols + i * BLOCK_SIZE + tIDx]; //Setting up sub array1 to contain relevant pieces of array1
}else {
subArray1[tIDy][tIDx] = 0; //0ing values to prevent miscalculation if not relevant
}
if (col < arr2_cols && i*BLOCK_SIZE+tIDy<arr1_cols) {
subArray2[tIDy][tIDx] = array2[(i * BLOCK_SIZE + tIDy)*arr2_cols+col]; //Setting up sub array2 to contain relevant pieces of array2
}else {
subArray2[tIDy][tIDx] = 0;//0ing values to prevent miscalculation if not relevant
}
__syncthreads(); //Blocking to ensure sub arrays are built
for (int ii = 0; ii < BLOCK_SIZE; ii++) {
result += subArray1[tIDy][ii] * subArray2[ii][tIDx]; //Calculating result for this chunk utilising many threads simultaneously
}
__syncthreads(); //Ensure result calculation is done
}
if (row < arr1_rows&&col < arr2_cols) {
output[row*arr2_cols + col] = result; //Calculate overall output in position
}

}