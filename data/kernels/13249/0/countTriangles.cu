#include "hip/hip_runtime.h"
#include "includes.h"
/**
*
* Copyright (C) Tyler Hackett 2016
*
* CUDA Triangle Counter
*
* A quickly-written program to determine all possible combinations of
* valid triangles from a grid, allowing for certain coordinates of the
* grid to be marked as unusable.
*
* main.cu
*
* */



__global__ void countTriangles(uint2 *validPoints, int *count)
{
/* Only allow operations on blocks where x < y < z, to prevent repeat triangles*/
if (blockIdx.x > blockIdx.y || blockIdx.y > blockIdx.z || blockIdx.x > blockIdx.z)
return;

uint2 x, y, z;
x = validPoints[blockIdx.x];
y = validPoints[blockIdx.y];
z = validPoints[blockIdx.z];

/*Check if the points are coplanar.*/
if ((x.x == y.x || x.y == y.y) && (y.x == z.x || y.y == z.y) && (x.x == z.x || x.y == z.y))
return;
/*Check for any coincident points.*/
if ((x.x == y.x && x.y == y.y) || (y.x == z.x && y.y == z.y) || (x.x == z.x && x.y == z.y))
return;

/*If the thread makes it this far, then we have a triangle that obeys the laws of geometry!*/
atomicAdd(count, 1);
}