#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid(float *inout, float *bias, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
float t = inout[i * cols + j];
inout[i * cols + j] = 1 / (1 + expf(-t)) + bias[i];
}