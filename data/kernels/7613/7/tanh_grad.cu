#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanh_grad(float *pre_grad, float *output, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
float t = output[i * cols + j];
pre_grad[i * cols + j] *= 1 - t * t;
}