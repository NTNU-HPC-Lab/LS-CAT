#include "hip/hip_runtime.h"
#include "includes.h"
/*
Modified from
https://github.com/zhxfl/CUDA-CNN
*/



__global__ void elementwiseMul(float *x, float *y, float *z, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
z[i * cols + j] = x[i * cols + j] * y[i * cols + j];
}