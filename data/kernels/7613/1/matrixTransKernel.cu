#include "hip/hip_runtime.h"
#include "includes.h"
/*
Modified from
https://github.com/zhxfl/CUDA-CNN
*/



__global__ void matrixTransKernel(float *A, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
float tmp = A[i * cols + j];
A[i * cols + j] = A[j * cols + i];
A[j * cols + i] = tmp;
}