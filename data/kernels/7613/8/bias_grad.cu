#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bias_grad(float *pre_grad, float *output, int rows, int cols) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= rows) return;
output[i] = 0;
for (int k = 0; k < cols; k++) {
output[i] += pre_grad[i * cols + k];
}
}