#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if((row < height) && (col < width)){
int pos = (row*width+col)*Channels;

imageOutput[row*width+col] = imageInput[pos+RED]*0.299 + imageInput[pos+GREEN]*0.587 + imageInput[pos+BLUE]*0.114;
}
}