#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if((i >= max_x) || (j >= max_y)) return;
int pixel_index = j*max_x + i;
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}