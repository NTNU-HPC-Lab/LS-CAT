#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bfsCheck( bool *d_graph_mask, bool *d_updating_graph_mask, bool *d_graph_visited, int no_of_nodes, bool *stop )
{
*stop = false;
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid < no_of_nodes){
if (d_updating_graph_mask[tid] == true){
d_graph_mask[tid] = true;
d_graph_visited[tid] = true;
*stop = true;
d_updating_graph_mask[tid] = false;
}
}
}