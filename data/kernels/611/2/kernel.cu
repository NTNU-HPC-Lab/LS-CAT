#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(const unsigned char * src, unsigned char * dst, int level, const size_t width, const size_t height)
{
const size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
const size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

if (xIndex < width && yIndex < height)
{
size_t o = yIndex * width + xIndex;
if (level == 256)
{
*(dst + o) = 0;
}
else
{
*(dst + o) = (*(src + o) >= level) ? 255 : 0;
}
// Notice how the below version avoids having an 'if' statement.
// I wonder if this is truly correct - I'll have to test this
// carefully someday but it works correctly. I figured the
// subtraction should cause an underflow which the shift might
// propagate through the rest of the byte so as to cause 255.
// *(dst + o) = ~((*(src + o) - level - 1) >> 7);
}
}