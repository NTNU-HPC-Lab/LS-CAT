#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vignette(const unsigned char * src, unsigned char * dst, float inner, float outer, const size_t width, const size_t height)
{
// the xIndex and yIndex will be used cordinates pixels of the image
// NOTE
// NOTE This assumes that we are treating this as a two dimensional data structure and the blocks will be used in the same way
// NOTE
size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

// Checking to see if the indexs are within the bounds of the image
if (xIndex < width && yIndex < height)
{
// offset represents postion of the current pixel in the one dimensional array
size_t offset = yIndex * width + xIndex;
// Shift the pixel oriented coordinates into image resolution independent coordinates
// where 0, 0 is the center of the image.
float x = xIndex / float(height) - float(width) / float(height) / 2.0f;
float y = yIndex / float(height) - 0.5f;
//Calculates current pixels distance from the center where the cordinates are 0, 0
float d = sqrtf(x * x + y * y);
if (d < inner)
{
// if d is less than inner boundary, we don't change that specific image pixel
*(dst + offset) = *(src + offset);
}
else if (d > outer)
{
// if d is greater than outer boundary, we set it to 0 so it becomes black
*(dst + offset) = 0;
}
else
{
// If in between the inner and outer boundaries, it will be a shade of gray.
// NOTE
// NOTE  This assumes... by the time we get here, we have checked that outer does not equal inner
// NOTE  This also assumes ... by the time we get here, we have made inner less than outer
// NOTE
float v = 1 - (d - inner) / (outer - inner);
*(dst + offset) = (unsigned char)(*(src + offset) * v);
}
}
}