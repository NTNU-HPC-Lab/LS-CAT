#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuSum(int *prices,int *sumpricesout,int days,int seconds,int N)
{
int currentday = blockIdx.x*blockDim.x + threadIdx.x;
if(currentday<days)
{
int start = currentday * seconds;
int end = start+seconds;

int totprice=0;
for(int j=start;j<end;++j)
totprice+=prices[j];

sumpricesout[currentday] = totprice;
}
}