#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void CycleRoutineGPU(char *CurrentState , char *NextState , int X , int Dimension){

int sum=0;
int id=blockIdx.x*blockDim.x + threadIdx.x;

if (id<Dimension) {

if(id+X<Dimension ) {
sum += CurrentState[id+X];
}
if(id-X>=0){
sum += CurrentState[id-X];
}
if(id/X == (id+1)/X) {
sum += CurrentState[id+1];
}
if(id/X == (id-1)/X) {
sum += CurrentState[id-1];
}
if(id+X<Dimension && (id+X)/X == (id+X+1)/X) {
sum += CurrentState[id+X+1];
}
if(id+X<Dimension && (id+X)/X == (id+X-1)/X) {
sum += CurrentState[id+X-1];
}
if(id-X>=0 && (id-X)/X == (id-X+1)/X) {
sum += CurrentState[id-X+1];
}
if(id-X>=0 && (id-X)/X == (id-X-1)/X) {
sum += CurrentState[id-X-1];
}


if (sum < 2 || sum > 3)
NextState[id] = 0;
else if (sum == 3)
NextState[id] =  1;
else
NextState[id] = CurrentState[id];

}

__syncthreads();
}