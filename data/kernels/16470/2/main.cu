#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "wireless_src_pulse_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int step = 1;
double amp = 1;
double MAX_TIME = 1;
double TIME_STEP = 1;
int radius = 1;
int source_active = 1;
int src_x = 1;
int src_y = 1;
double *ua_gpu = NULL;
hipMalloc(&ua_gpu, XSIZE*YSIZE);
double *ub_gpu = NULL;
hipMalloc(&ub_gpu, XSIZE*YSIZE);
double *uc_gpu = NULL;
hipMalloc(&uc_gpu, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
wireless_src_pulse_kernel<<<gridBlock,threadBlock>>>(step,amp,MAX_TIME,TIME_STEP,radius,source_active,src_x,src_y,ua_gpu,ub_gpu,uc_gpu);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
wireless_src_pulse_kernel<<<gridBlock,threadBlock>>>(step,amp,MAX_TIME,TIME_STEP,radius,source_active,src_x,src_y,ua_gpu,ub_gpu,uc_gpu);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
wireless_src_pulse_kernel<<<gridBlock,threadBlock>>>(step,amp,MAX_TIME,TIME_STEP,radius,source_active,src_x,src_y,ua_gpu,ub_gpu,uc_gpu);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}