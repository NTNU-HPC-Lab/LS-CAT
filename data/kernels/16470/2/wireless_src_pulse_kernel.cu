#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int is_source_gpu(int i, int j, int radius, int source_active, int src_x, int src_y)
{
if (!source_active)
return 0;
if (sqrt(pow((float)(src_x - i), 2) + pow((float)(src_y - j), 2)) <= radius)
return 1;
return 0;
}
__global__ void wireless_src_pulse_kernel(int step, double amp, double MAX_TIME, double TIME_STEP, int radius, int source_active, int src_x, int src_y, double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
int i, j;
int i_start, j_start;
int i_final, j_final;
int line_length;
int global_thread_x, global_thread_y;
int thread_work = 32;

line_length = gridDim.y * blockDim.y;

global_thread_x = blockDim.x * blockIdx.x + threadIdx.x;
global_thread_y = blockDim.y * blockIdx.y + threadIdx.y;

i_start	= global_thread_x * thread_work;
j_start	= global_thread_y * thread_work;
i_final	= global_thread_x * (thread_work + 1);
j_final = global_thread_y * (thread_work + 1);

if (step < (int)(MAX_TIME / TIME_STEP) / 2){
// Pulse source
for (i = i_start; i < i_final; i++){
for (j = j_start; j < j_final; j++){
if (is_source_gpu(i, j, radius, 1, src_x, src_y))
uc_gpu[i * line_length + j] = amp * fabs(sin(step * M_PI/4));
}
}
} else if (source_active){
for (i = i_start; i < i_final; i++) {
for (j = j_start; j < j_final; j++) {
if (is_source_gpu(i, j, radius, source_active, src_x, src_y)) {
ua_gpu[i * line_length + j] = 0;
ub_gpu[i * line_length + j] = 0;
uc_gpu[i * line_length + j] = 0;
}
}
}
}
// All threads should reach this point before setting source_active.
// Option 1:  need a thread barrier here -> not done, I chose option 2
// Option 2:  simply write 2 kernels and syncCPU -> done, I chose this option
//	 	CPU is setting source_active = 0 after this kernel is done executing.
}