#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_all_zero_kernel(double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
ua_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
ub_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
// TODO: sync CPU after this -> move to utils.cu file
}