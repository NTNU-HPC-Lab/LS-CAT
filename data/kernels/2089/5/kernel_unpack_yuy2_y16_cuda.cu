#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

if (i >= superPixCount)
return;

for (; i < superPixCount; i += stride) {

int idx = i * 4;

dst[idx] = 0;
dst[idx + 1] = src[idx + 0];
dst[idx + 2] = 0;
dst[idx + 3] = src[idx + 2];
}
}