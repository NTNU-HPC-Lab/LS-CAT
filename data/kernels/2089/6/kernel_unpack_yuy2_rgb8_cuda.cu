#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

if (i >= superPixCount)
return;

for (; i < superPixCount; i += stride) {

int idx = i * 4;

uint8_t y0 = src[idx];
uint8_t u0 = src[idx + 1];
uint8_t y1 = src[idx + 2];
uint8_t v0 = src[idx + 3];

int16_t c = y0 - 16;
int16_t d = u0 - 128;
int16_t e = v0 - 128;

int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

int odx = i * 6;

dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);

c = y1 - 16;

dst[odx + 3] = clamp((298 * c + 409 * e + 128) >> 8);
dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
dst[odx + 5] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp

}
}