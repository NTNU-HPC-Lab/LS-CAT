#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculation(    char *a, char *b, int *c, int constant, int vector_size ) {

int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

__shared__ char sharedDataA[block_size+2]; // border for the block are needed
char curr_b;

// Populate border
if (threadIdx.x == 0){
sharedDataA[0] = a[tid>0?tid-1:(vector_size-1)];

} else if (threadIdx.x == block_size - 1){
sharedDataA[block_size + 1] = a[tid<(vector_size-1)?tid+1:0];

} else if (tid == vector_size - 1){
sharedDataA[threadIdx.x + 2] = a[0];
}

// How can we avoid these ifs??? Tip: Padding
if (tid < vector_size){
// Populate shared data for A
sharedDataA[threadIdx.x+1] = a[tid];

// Bring data from B (no need for shared)
curr_b = b[tid];
}

__syncthreads();

// Perform calculation
if (tid < vector_size){
int output_c = (sharedDataA[threadIdx.x]-sharedDataA[threadIdx.x+2])*curr_b; //Use neighbors from shared data
output_c += sharedDataA[threadIdx.x+1]*constant;

// Write result
c[tid] = output_c;
}
}