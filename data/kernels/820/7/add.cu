#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
int i= blockIdx.x*blockDim.x+threadIdx.x;

c[i]= a[i]+b[i];

}