#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < numOfPrimes) {
for(int i = d_PL[numOfPrimes-1]+1; i < lenInputList; i++) {
if(i % d_PL[index] == 0) {
d_IL[i] = 0;
}
}
}
}