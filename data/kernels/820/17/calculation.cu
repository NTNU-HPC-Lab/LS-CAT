#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculation(    char* dev_a, char* dev_b, char* dev_c, int num_matrices, int matrix_size ) {
// Each thread handles a matrix
int k = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (k >= num_matrices) return;

// If first element is different than 0 do the computation
if (dev_a[k*matrix_size*matrix_size] != 0){
for (int j = 0; j < matrix_size; j++){
//If first value in the row of the matrix, do addition
if (dev_a[k*matrix_size*matrix_size+j*matrix_size] < threshold){
for (int i = 0; i < matrix_size; i++){
int index = k*matrix_size*matrix_size+j*matrix_size+i;
dev_c[index] = dev_a[index] + dev_b[index];
}
//Do subtraction
} else {
for (int i = 0; i < matrix_size; i++){
int index = k*matrix_size*matrix_size+j*matrix_size+i;
dev_c[index] = dev_a[index] - dev_b[index];
}
}
}
}
}