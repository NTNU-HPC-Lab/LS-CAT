#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startInputlist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{

uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

//uint64_cu primes= d_prime_list[tid];
/*if(tid< d_number_of_primes[0])
printf("%d ---->  %llu\n",tid,primes);*/

//printf("THE NUMBER OF PRIMES ARE: %llu\n",*d_number_of_primes);
if (tid < *d_total_inputsize) {
//printf("Kaustubh\n");


uint64_cu actualNumber=*d_startInputlist+tid;
for(uint64_cu i=0;i<*d_number_of_primes;i++) { // Added less than eual to here.
uint64_cu bucket= tid/(WORD);
int setbit= tid%(WORD);

if(actualNumber%d_prime_list[i]==0) {
//printf("%llu is divisible by %llu \n", number,primes);
// THIS WAS WRONG  : d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
atomicOr(&d_input_list[bucket],1U<<setbit); // setbit
break;
}
}
}
}