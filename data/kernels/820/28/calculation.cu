#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculation(    int *a, int *b, int *c, int constant, int vector_size ) {

int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (tid < vector_size){

// Read in inputs
int prev_a = a[tid>0?tid-1:(vector_size-1)];
int curr_a = a[tid];
int post_a = a[tid<(vector_size-1)?tid+1:0];

int curr_b = b[tid];

// Do computation
int output_c = (prev_a-post_a)*curr_b + curr_a*constant;

// Write result
c[tid] = output_c;
}
}