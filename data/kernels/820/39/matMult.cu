#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMult(int* a, int* b, int* res,unsigned  int rows, unsigned int k, unsigned int cols){
unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;

unsigned int sum = 0;

if(r< rows && c< cols){
for(int x=0; x<k; x++){
sum += a[r*k +x] + b[x*cols + c];
}
res[r*cols + c] = sum;
}
}