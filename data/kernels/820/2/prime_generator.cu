#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{

uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

//uint64_cu primes= d_prime_list[tid];
/*if(tid< d_number_of_primes[0])
printf("%d ---->  %llu\n",tid,primes);*/

//printf("THE NUMBER OF PRIMES ARE: %llu\n",*d_number_of_primes);
if (tid < *d_number_of_primes) {
//printf("Kaustubh\n");
uint64_cu primes=d_prime_list[tid];
for(uint64_cu i=0;i<d_total_inputsize[0];i++) { // Added less than eual to here.
uint64_cu bucket= i/(WORD);
int setbit= i%(WORD);
uint64_cu number=d_startPrimelist[0]+i;
//printf("THE NUMBER %llu IS BEING DIVIDED BY %llu\n",number,primes);
if(number%primes==0) {
//printf("%llu is divisible by %llu \n", number,primes);
// THIS WAS WRONG  : d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
if(0 == (d_input_list[bucket] & 1U<<setbit)){ // testbit
atomicOr(&d_input_list[bucket],1U<<setbit); // setbit
}
}
}
}
}