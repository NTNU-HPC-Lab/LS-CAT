#include "hip/hip_runtime.h"
#include "includes.h"

#define block_size   32
#define vector_size  10000000


__global__ void add( int *a, int *b, int *c ) {
int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (tid < vector_size){
c[tid] = a[tid] + b[tid];                   // add vectors together
}
}