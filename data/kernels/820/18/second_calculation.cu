#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void second_calculation( char* dev_a, char* dev_b, char* dev_c, int k, int num_matrices, int matrix_size ) {
// Each thread handles a matrix
int j = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (j >= matrix_size) return;

//If first value in the row of the matrix, do addition
if (dev_a[k*matrix_size*matrix_size+j*matrix_size] < threshold){
for (int i = 0; i < matrix_size; i++){
int index = k*matrix_size*matrix_size+j*matrix_size+i;
dev_c[index] = dev_a[index] + dev_b[index];
}
//Do subtraction
} else {
for (int i = 0; i < matrix_size; i++){
int index = k*matrix_size*matrix_size+j*matrix_size+i;
dev_c[index] = dev_a[index] - dev_b[index];
}
}
}