#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{

uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

if (tid < *d_number_of_primes) {
//            printf("Kaustubh\n");
uint64_cu primes=d_prime_list[tid];
//  printf("%llu\n",primes);
for(uint64_cu i=0;i<=d_total_inputsize[0];i++) // Added less than eual to here.
{
uint64_cu bucket= i/(WORD);
uint64_cu setbit= i%(WORD);
uint64_cu number=d_startPrimelist[0]+i;

//      printf("%llu -----> hash the value %llu to %llu bucket and change the %llu bit\n",number,i,bucket,setbit );
//      printf("**************  %llu --- %llu \n",number,primes);
if(number%primes==0)
{
//                                        printf("%llu is divisible by %llu \n", number,primes);
d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
}
}
}
}