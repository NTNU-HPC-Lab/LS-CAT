#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_calculation(    char* dev_a, char* dev_b, char* dev_c, int k, int j, int num_matrices, int matrix_size ) {
// Each thread handles a matrix
int i = (blockIdx.x*blockDim.x) + threadIdx.x;

if (i >= matrix_size) return;

int index = k*matrix_size*matrix_size+j*matrix_size+i;
dev_c[index] = dev_a[index] - dev_b[index];

}