#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculation(    char *a, char *b, int *c, int constant, int vector_size ) {

int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (tid < vector_size){

// Read in inputs
char prev_a = a[tid>0?tid-1:(vector_size-1)];
char curr_a = a[tid];
char post_a = a[tid<(vector_size-1)?tid+1:0];

char curr_b = b[tid];

// Do computation
int output_c = (prev_a-post_a)*curr_b + curr_a*constant;

// Write result
c[tid] = output_c;
}
}