#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

if (tid < vector_size){
c[tid] = a[tid] + b[tid];                   // add vectors together
}
}