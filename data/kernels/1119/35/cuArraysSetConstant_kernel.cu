#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysSetConstant_kernel(float *image, int size, float value)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;

if(idx < size)
{
image[idx] = value;
}
}