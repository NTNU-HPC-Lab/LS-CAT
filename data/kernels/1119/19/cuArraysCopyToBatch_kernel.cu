#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyToBatch_kernel(const float2 *imageIn, const int inNX, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImagesX, const int nImagesY, const int strideX, const int strideY)
{
int idxImage = blockIdx.z;
int outx = threadIdx.x + blockDim.x*blockIdx.x;
int outy = threadIdx.y + blockDim.y*blockIdx.y;
if(idxImage >=nImagesX*nImagesY|| outx >= outNX || outy >= outNY) return;
int idxOut = idxImage*outNX*outNY + outx*outNY + outy;
int idxImageX = idxImage/nImagesY;
int idxImageY = idxImage%nImagesY;
int idxIn = (idxImageX*strideX+outx)*inNY + idxImageY*strideY+outy;
imageOut[idxOut] = imageIn[idxIn];
}