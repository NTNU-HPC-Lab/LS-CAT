#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyExtractVaryingOffsetCorr(const float *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, int *imageValid, const int nImages, const int2 *maxloc)
{

int idxImage = blockIdx.z;

int outx = threadIdx.x + blockDim.x*blockIdx.x;
int outy = threadIdx.y + blockDim.y*blockIdx.y;

int inx = outx + maxloc[idxImage].x - outNX/2;
int iny = outy + maxloc[idxImage].y - outNY/2;

if (outx < outNX && outy < outNY)
{
int idxOut = ( blockIdx.z * outNX + outx ) * outNY + outy;

int idxIn = ( blockIdx.z * inNX + inx ) * inNY + iny;

if (inx>=0 && iny>=0 && inx<inNX && iny<inNY) {

imageOut[idxOut] = imageIn[idxIn];
imageValid[idxOut] = 1;
}
else {
imageOut[idxOut] = 0.0f;
imageValid[idxOut] = 0;
}
}
}