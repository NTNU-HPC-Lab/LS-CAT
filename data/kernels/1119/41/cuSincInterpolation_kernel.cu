#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSincInterpolation_kernel(const int nImages, const float * imagesIn, const int inNX, const int inNY, float * imagesOut, const int outNX, const int outNY, const float * r_filter_, const int i_covs_, const int i_decfactor_, const int i_intplength_, const int i_startX, const int i_startY, const int i_int_size)
{
int idxImage = blockIdx.z;
int idxX = threadIdx.x + blockDim.x*blockIdx.x;
int idxY = threadIdx.y + blockDim.y*blockIdx.y;
if(idxImage >=nImages || idxX >= i_int_size || idxY >= i_int_size) return;
int outx = idxX + i_startX;
int outy = idxY + i_startY;
int idxOut = idxImage*outNX*outNY + outx*outNY + outy;

float r_xout = (float)outx/i_covs_;
int i_xout = int(r_xout);
float r_xfrac = r_xout - i_xout;
int i_xfrac = int(r_xfrac*i_decfactor_);

float r_yout = (float)outy/i_covs_;
int i_yout = int(r_yout);
float r_yfrac = r_yout - i_yout;
int i_yfrac = int(r_yfrac*i_decfactor_);

float intpData = 0.0f;
float r_sincwgt = 0.0f;
float r_sinc_coef;

for(int i=0; i < inNX; i++) {
int i_xindex = i_xout - i + i_intplength_/2;
if(i_xindex < 0) i_xindex+= i_intplength_;
if(i_xindex >= i_intplength_) i_xindex-=i_intplength_;
float r_xsinc_coef = r_filter_[i_xindex*i_decfactor_+i_xfrac];

for(int j=0; j< inNY; j++) {
int i_yindex = i_yout - j + i_intplength_/2;
if(i_yindex < 0) i_yindex+= i_intplength_;
if(i_yindex >= i_intplength_) i_yindex-=i_intplength_;
float r_ysinc_coef = r_filter_[i_yindex*i_decfactor_+i_yfrac];
r_sinc_coef = r_xsinc_coef*r_ysinc_coef;
r_sincwgt += r_sinc_coef;
intpData += imagesIn[idxImage*inNX*inNY+i*inNY+j]*r_sinc_coef;
/*
if(outx == 0 && outy == 1) {
printf("intp kernel %d %d %d %d %d %d %d %f\n", i, j, i_xindex, i_yindex, i_xindex*i_decfactor_+i_xfrac,
i_yindex*i_decfactor_+i_yfrac, idxImage*inNX*inNY+i*inNY+j, r_sinc_coef);
}*/
}
}
imagesOut[idxOut] = intpData/r_sincwgt;
//printf("test int kernel %d %d %f %f %f\n", outx, outy, intpData, r_sincwgt, imagesOut[idxOut]);
}