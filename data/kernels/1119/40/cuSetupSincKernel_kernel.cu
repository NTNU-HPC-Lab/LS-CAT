#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSetupSincKernel_kernel(float *r_filter_, const int i_filtercoef_, const float r_soff_, const float r_wgthgt_, const int i_weight_, const float r_soff_inverse_, const float r_beta_, const float r_decfactor_inverse_, const float r_relfiltlen_inverse_)
{
int i = threadIdx.x + blockDim.x*blockIdx.x;
if(i > i_filtercoef_) return;
float r_wa = i - r_soff_;
float r_wgt = (1.0f - r_wgthgt_) + r_wgthgt_*cos(PI*r_wa*r_soff_inverse_);
float r_s = r_wa*r_beta_*r_decfactor_inverse_*PI;
float r_fct;
if(r_s != 0.0f) {
r_fct = sin(r_s)/r_s;
}
else {
r_fct = 1.0f;
}
if(i_weight_ == 1) {
r_filter_[i] = r_fct*r_wgt;
}
else {
r_filter_[i] = r_fct;
}
//printf("kernel %d %f\n", i, r_filter_[i]);
}