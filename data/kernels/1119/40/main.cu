#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cuSetupSincKernel_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *r_filter_ = NULL;
hipMalloc(&r_filter_, XSIZE*YSIZE);
const int i_filtercoef_ = 1;
const float r_soff_ = 1;
const float r_wgthgt_ = 1;
const int i_weight_ = 1;
const float r_soff_inverse_ = 1;
const float r_beta_ = 1;
const float r_decfactor_inverse_ = 1;
const float r_relfiltlen_inverse_ = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cuSetupSincKernel_kernel<<<gridBlock,threadBlock>>>(r_filter_,i_filtercoef_,r_soff_,r_wgthgt_,i_weight_,r_soff_inverse_,r_beta_,r_decfactor_inverse_,r_relfiltlen_inverse_);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cuSetupSincKernel_kernel<<<gridBlock,threadBlock>>>(r_filter_,i_filtercoef_,r_soff_,r_wgthgt_,i_weight_,r_soff_inverse_,r_beta_,r_decfactor_inverse_,r_relfiltlen_inverse_);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cuSetupSincKernel_kernel<<<gridBlock,threadBlock>>>(r_filter_,i_filtercoef_,r_soff_,r_wgthgt_,i_weight_,r_soff_inverse_,r_beta_,r_decfactor_inverse_,r_relfiltlen_inverse_);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}