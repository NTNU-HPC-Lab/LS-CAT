#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernel_maxlocPlusZoominOffset(float *offset, const int * padStart, const int * maxlocUpSample, const size_t nImages, float zoomInRatioX, float zoomInRatioY)
{
int imageIndex = threadIdx.x + blockDim.x *blockIdx.x; //image index
if (imageIndex < nImages)
{
int index=2*imageIndex;
offset[index] = padStart[index] + maxlocUpSample[index] * zoomInRatioX;
index++;
offset[index] = padStart[index] + maxlocUpSample[index] * zoomInRatioY;
}
}