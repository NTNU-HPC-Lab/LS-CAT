#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyExtractVaryingOffset_C2C(const float2 *imageIn, const int inNX, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImages, const int2 *offsets)
{
int outx = threadIdx.x + blockDim.x*blockIdx.x;
int outy = threadIdx.y + blockDim.y*blockIdx.y;

if(outx < outNX && outy < outNY)
{
int idxImage = blockIdx.z;
int idxOut = (blockIdx.z * outNX + outx)*outNY+outy;
int idxIn = (blockIdx.z*inNX + outx + offsets[idxImage].x)*inNY + outy + offsets[idxImage].y;
imageOut[idxOut] = imageIn[idxIn];
}
}