#include "hip/hip_runtime.h"
#include "includes.h"
/*
* cuArraysPadding.cu
* Padding Utitilies for oversampling
*/


//padding zeros in the middle, move quads to corners
//for raw chunk data oversampling
//tested
__global__ void cuArraysR2C_kernel(float *image1, float2 *image2, int size)
{
int idx =  threadIdx.x + blockDim.x*blockIdx.x;
if(idx < size)
{
image2[idx].x = image1[idx];
image2[idx].y =  0.0f;
}
}