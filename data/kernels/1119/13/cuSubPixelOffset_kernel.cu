#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSubPixelOffset_kernel(const int2 *offsetInit, const int2 *offsetZoomIn, float2 *offsetFinal, const float OSratio, const float xoffset, const float yoffset, const int size)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;
if (idx >= size) return;
offsetFinal[idx].x = OSratio*(offsetZoomIn[idx].x ) + offsetInit[idx].x  - xoffset;
offsetFinal[idx].y = OSratio*(offsetZoomIn[idx].y ) + offsetInit[idx].y - yoffset;
}