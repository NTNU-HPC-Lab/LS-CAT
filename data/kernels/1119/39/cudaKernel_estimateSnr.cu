#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernel_estimateSnr(const float* corrSum, const int* corrValidCount, const float* maxval, float* snrValue, const int size)

{
int idx = threadIdx.x + blockDim.x*blockIdx.x;

if (idx >= size) return;

float mean = (corrSum[idx] - maxval[idx] * maxval[idx]) / (corrValidCount[idx] - 1);

snrValue[idx] = maxval[idx] / mean;
}