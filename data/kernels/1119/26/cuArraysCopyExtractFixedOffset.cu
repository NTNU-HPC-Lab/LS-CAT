#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyExtractFixedOffset(const float *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, const int nImages, const int offsetX, const int offsetY)
{
int outx = threadIdx.x + blockDim.x*blockIdx.x;
int outy = threadIdx.y + blockDim.y*blockIdx.y;

if(outx < outNX && outy < outNY)
{
int idxOut = (blockIdx.z * outNX + outx)*outNY+outy;
int idxIn = (blockIdx.z*inNX + outx + offsetX)*inNY + outy + offsetY;
imageOut[idxOut] = imageIn[idxIn];
}
}