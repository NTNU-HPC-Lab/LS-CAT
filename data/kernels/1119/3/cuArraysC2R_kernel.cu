#include "hip/hip_runtime.h"
#include "includes.h"
/*
* cuArraysPadding.cu
* Padding Utitilies for oversampling
*/


//padding zeros in the middle, move quads to corners
//for raw chunk data oversampling
//tested
__global__ void cuArraysC2R_kernel(float2 *image1, float *image2, int size)
{
int idx =  threadIdx.x + blockDim.x*blockIdx.x;
if(idx < size)
{
image2[idx] = image1[idx].x;
}
}