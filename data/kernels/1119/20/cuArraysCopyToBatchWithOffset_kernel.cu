#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyToBatchWithOffset_kernel(const float2 *imageIn, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImages, const int *offsetX, const int *offsetY)
{
int idxImage = blockIdx.z;
int outx = threadIdx.x + blockDim.x*blockIdx.x;
int outy = threadIdx.y + blockDim.y*blockIdx.y;
if(idxImage>=nImages || outx >= outNX || outy >= outNY) return;
int idxOut = idxImage*outNX*outNY + outx*outNY + outy;
int idxIn = (offsetX[idxImage]+outx)*inNY + offsetY[idxImage] + outy;
imageOut[idxOut] = imageIn[idxIn];
}