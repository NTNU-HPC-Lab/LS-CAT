#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_s(unsigned int * ind, const size_t nbn, const unsigned int ne)
{
int m_i_b = threadIdx.x;
if (m_i_b >= ne)  return;
extern __shared__ float dats[];
dats[m_i_b] = ind[m_i_b];
__syncthreads();

for (int q = 1; q < nbn; q *= 2) {
if (m_i_b >= q) {
dats[m_i_b] += dats[m_i_b - q];
}
__syncthreads();
}
if (m_i_b == 0)  ind[0] = 0;
else  ind[m_i_b] = dats[m_i_b - 1];
}