#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_m(unsigned int * ind, unsigned int *scand, unsigned int shift, const unsigned int ne)
{
unsigned int sosm = 1 << shift;
int m_i_b = threadIdx.x + blockDim.x * blockIdx.x;
if (m_i_b >= ne)  return;
scand[m_i_b] = ((ind[m_i_b] & sosm) >> shift) ? 0 : 1;
}