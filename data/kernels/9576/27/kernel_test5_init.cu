#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned int*) end_ptr) {
return;
}

unsigned int p1 = 1;
for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i+=16){
unsigned int p2 = ~p1;

ptr[i] = p1;
ptr[i+1] = p1;
ptr[i+2] = p2;
ptr[i+3] = p2;
ptr[i+4] = p1;
ptr[i+5] = p1;
ptr[i+6] = p2;
ptr[i+7] = p2;
ptr[i+8] = p1;
ptr[i+9] = p1;
ptr[i+10] = p2;
ptr[i+11] = p2;
ptr[i+12] = p1;
ptr[i+13] = p1;
ptr[i+14] = p2;
ptr[i+15] = p2;

p1 = p1<<1;
if (p1 == 0){
p1 = 1;
}
}

return;
}