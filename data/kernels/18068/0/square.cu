#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void square(float* d_in, float* d_out) {
int idx =  threadIdx.x;
float f = d_in[idx];
d_out[idx] = f * f;
}