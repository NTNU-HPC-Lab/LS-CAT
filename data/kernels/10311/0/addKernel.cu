#include "hip/hip_runtime.h"
#include "includes.h"
int addWithCuda(int *c, const int *a, const int *b, unsigned int size);
bool isLoadDevice = false;



// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}