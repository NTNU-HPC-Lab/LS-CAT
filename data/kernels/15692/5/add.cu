#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
//blockDim is num threads/block, multiplied by block number to index to one of them, then select thread inside block via thread Id
int threadID = threadIdx.x + blockIdx.x * blockDim.x;
//Max 65 535 blocks, with 512 threads each ~ 8 million elements, if vector exceeds that amount require a soln
//Run arbitrary number of blocks and threads
//Done at each parallel process, allows a single launch of threads to iteratively cycle through all available indices of vector
//As long as each thread begins at a unique index-val, all will iterate arr without affecting one another
while (threadID < N)
{
c[threadID] = a[threadID] + b[threadID];
//Add
threadID += blockDim.x * gridDim.x;
}
}