#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(unsigned char *ptr, int ticks)
{
//Index one of the threads to an image pos
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

float fx = x - DIM / 2;
float fy = y - DIM / 2;
float d = sqrtf(fx * fx + fy * fy);
//Create varying grey vals depending on pixel
unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

//Offset into output buffer for window generation when ready
ptr[offset * 4 + 0] = grey;
ptr[offset * 4 + 1] = grey;
ptr[offset * 4 + 2] = grey;
ptr[offset * 4 + 3] = 255;
}