#include "hip/hip_runtime.h"
#include "includes.h"
//********************************************************//
// CUDA SIFT extractor by Marten Bjorkman aka Celebrandil //
//********************************************************//


///////////////////////////////////////////////////////////////////////////////
// Kernel configuration
///////////////////////////////////////////////////////////////////////////////

__constant__ float d_Threshold[2];
__constant__ float d_Scales[8], d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;

__device__ unsigned int d_PointCounter[1];
__constant__ float d_Kernel1[5];
__constant__ float d_Kernel2[12*16];

///////////////////////////////////////////////////////////////////////////////
// Lowpass filter an subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDown_D(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch) {
// TODO: one element per thread in a block?
__shared__ float inrow[SCALEDOWN_W + 4];

__shared__ float brow[5 * (SCALEDOWN_W / 2)];

//
__shared__ int yRead[SCALEDOWN_H + 4];
__shared__ int yWrite[SCALEDOWN_H + 4];

// Get thread index, which ranges from 0 to SCALEDOWN_W + 4
const int tx = threadIdx.x;

// Get indices in brow
// TODO: move this out?
#define dx2 (SCALEDOWN_W / 2)
const int tx0 = tx + 0 * dx2;
const int tx1 = tx + 1 * dx2;
const int tx2 = tx + 2 * dx2;
const int tx3 = tx + 3 * dx2;
const int tx4 = tx + 4 * dx2;

// TODO: x and y pixel index
const int xStart = blockIdx.x * SCALEDOWN_W;
const int yStart = blockIdx.y * SCALEDOWN_H;

// TODO: x coordinate to write to?
const int xWrite = xStart / 2 + tx;
int xRead = xStart + tx - 2;
xRead = (xRead < 0 ? 0 : xRead);
xRead = (xRead >= width ? width - 1 : xRead);

const float *k = d_Kernel1;

// Identify y read and write indices; note we ignore SCALEDOWN_H + 4 <= tx <
// SCALEDOWN_H + 4 in this section
if (tx < SCALEDOWN_H + 4) {
// TODO: tx = 0 and tx = 1 are the same; why?
int y = yStart + tx - 1;

// Clamp at 0 and height - 1
y = (y < 0 ? 0 : y);
y = (y >= height ? height - 1 : y);

// Read start index
yRead[tx] = y * pitch;

// Write start index
yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
}

// Synchronize threads to ensure we have yRead and yWrite filled for current
// warp
__syncthreads();

// For each thread (which runs 0 to SCALEDOWN_W + 4 - 1), loop through 0 to
// SCALEDOWN_H + 4 - 1 by kernel size.
for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {

// yRead[dy + 0] is the y index to 0th row of data from source image (may
// be the same as 1st, 2nd, etc row, depending on how close we are to the
// edge of image). xRead is determined by thread id and starts from size
// of kernel / 2 + 1 to the left of our current pixel
inrow[tx] = d_Data[yRead[dy + 0] + xRead];

// Once we synchronize, inrow should contain the data from the source
// image corresponding to the first row in the current block. It is length
// SCALEDOWN_W + 4.
__syncthreads();

// For the SCALEDOWN_W / 2 threads in block, compute the first of 5
// indices for this thread. Convolve the 1-D kernel k with every other
// 'pixel' in the block via 2 * tx
if (tx < dx2) {
brow[tx0] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) +
k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) +
k[2] * inrow[2 * tx + 2];
}

// TODO: Once we synchronize, brow[tx0] should contain
__syncthreads();

// Compute for SCALEDOWN_W / 2 threads in block. dy & 1 is true if dy is
// odd. We require that dy is even and after we've completed at least one
// iteration
if (tx < dx2 && dy >= 4 && !(dy & 1)) {
d_Result[yWrite[dy + 0] + xWrite] = k[2] * brow[tx2] +
k[0] * (brow[tx0] + brow[tx4]) +
k[1] * (brow[tx1] + brow[tx3]);
}

// And...this is all just the same as above. One big unrolled for loop.
if (dy < (SCALEDOWN_H + 3)) {
// yRead[dy + 1] is the y index to 1th row of data from source image
// (may be the same as 1st, 2nd, etc row, depending on how close we are
// to the edge of image). xRead is determined by thread id and starts
// from size of kernel / 2 + 1 to the left of our current pixel
inrow[tx] = d_Data[yRead[dy + 1] + xRead];

__syncthreads();
if (tx < dx2) {
brow[tx1] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) +
k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) +
k[2] * inrow[2 * tx + 2];
}
__syncthreads();
if (tx<dx2 && dy>=3 && (dy&1)) {
d_Result[yWrite[dy+1] + xWrite] = k[2]*brow[tx3] + k[0]*(brow[tx1]+brow[tx0]) + k[1]*(brow[tx2]+brow[tx4]);
}
}
if (dy<(SCALEDOWN_H+2)) {
inrow[tx] = d_Data[yRead[dy+2] + xRead];
__syncthreads();
if (tx<dx2) {
brow[tx2] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
}
__syncthreads();
if (tx<dx2 && dy>=2 && !(dy&1)) {
d_Result[yWrite[dy+2] + xWrite] = k[2]*brow[tx4] + k[0]*(brow[tx2]+brow[tx1]) + k[1]*(brow[tx3]+brow[tx0]);
}
}
if (dy<(SCALEDOWN_H+1)) {
inrow[tx] = d_Data[yRead[dy+3] + xRead];
__syncthreads();
if (tx<dx2) {
brow[tx3] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
}
__syncthreads();
if (tx<dx2 && dy>=1 && (dy&1)) {
d_Result[yWrite[dy+3] + xWrite] = k[2]*brow[tx0] + k[0]*(brow[tx3]+brow[tx2]) + k[1]*(brow[tx4]+brow[tx1]);
}
}
if (dy<SCALEDOWN_H) {
inrow[tx] = d_Data[yRead[dy+4] + xRead];
__syncthreads();
if (tx<dx2) {
brow[tx4] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
}
__syncthreads();
if (tx<dx2 && !(dy&1)) {
d_Result[yWrite[dy+4] + xWrite] = k[2]*brow[tx1] + k[0]*(brow[tx4]+brow[tx3]) + k[1]*(brow[tx0]+brow[tx2]);
}
}
__syncthreads();
}
}