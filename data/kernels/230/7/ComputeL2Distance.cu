#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeL2Distance(float *corrData, int numPts1) {
// Get the global point index, not the local index within our 16x16 chunk
const int p1 = blockIdx.x * 16 + threadIdx.x;
const int p2 = blockIdx.y * 16 + threadIdx.y;

// Make sure p1 and p2 are both within bounds
if (p1 < numPts1) {
const int idx = p1 * gridDim.y * 16 + p2;
if (corrData[idx] > -1) corrData[idx] = 2 - 2 * corrData[idx];
else corrData[idx] = FLT_MAX;
}
}