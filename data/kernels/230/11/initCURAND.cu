#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initCURAND(unsigned int seed, hiprandState_t* d_states) {

/* we have to initialize the state */
hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
blockIdx.x, /* the sequence number should be different for each core (unless you want all
cores to get the same sequence of numbers for some reason - use thread id! */
0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&d_states[blockIdx.x]);
}