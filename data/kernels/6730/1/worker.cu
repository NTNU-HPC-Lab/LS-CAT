#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void worker(double * a, long n) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < n) {
a[i] += i;
}
}