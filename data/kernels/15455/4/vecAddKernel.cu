#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAddKernel(float *a, float *b, float *c, int n)
{
//ID del thread
int id = blockIdx.x*blockDim.x+threadIdx.x;


//No salir del tamaño del vector
if (id < n)
c[id] = a[id] + b[id];
}