#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16



__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

if( Col < Width && Row < Width)
{
float Pvalue = 0;
for(int k = 0; k < Width; ++k)
{
Pvalue += M[Row * Width + k] * N[k * Width + Col];
}
P[Row * Width + Col] = Pvalue;
}
}