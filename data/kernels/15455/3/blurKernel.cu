#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blurKernel(uchar3 *in, uchar3 *out, int w, int h)
{
int Col = blockIdx.x*blockDim.x + threadIdx.x;
int Row = blockIdx.y*blockDim.y + threadIdx.y;

if(Col<w && Row<h)
{
int pixVal1 = 0;
// int pixVal2 = 0;
// int pixVal3 = 0;
int	pixels1 = 0;
// int pixels2 = 0;
// int pixels3 = 0;

for(int blurRow=-BLUR_SIZE; blurRow<BLUR_SIZE+1;++blurRow){
for(int blurCol=-BLUR_SIZE; blurCol<BLUR_SIZE+1;++blurCol)
{
int curRow = Row + blurRow;
int curCol = Col + blurCol;

if(curRow>-1 && curRow<h && curCol>-1 && curCol<w){
pixVal1+=static_cast<int>(in[curRow*w + curCol].x);
pixels1++;
pixVal1+=static_cast<int>(in[curRow*w + curCol].y);
pixels1++;
pixVal1+=static_cast<int>(in[curRow*w + curCol].z);
pixels1++;

}
}

}

out[Row*w+Col].x= static_cast<unsigned char>(pixVal1/pixels1);
out[Row*w+Col].y= static_cast<unsigned char>(pixVal1/pixels1);
out[Row*w+Col].z= static_cast<unsigned char>(pixVal1/pixels1);

}
}