#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void color_to_grey(uchar3 *input_image, uchar3 *output_image, int width, int height)
{
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

if(col < width && row < height)
{
int pos = row * width + col;
output_image[pos].x = static_cast<unsigned char>(input_image[pos].x * 0.2126f + input_image[pos].y * 0.7125f + input_image[pos].z * 0.0722f);
output_image[pos].y = static_cast<unsigned char>(input_image[pos].x * 0.2126f + input_image[pos].y * 0.7125f + input_image[pos].z * 0.0722f);
output_image[pos].z = static_cast<unsigned char>(input_image[pos].x * 0.2126f + input_image[pos].y * 0.7125f + input_image[pos].z * 0.0722f);
}
}