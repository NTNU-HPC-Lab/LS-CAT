#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *a, float *b, float *c) {
int tid = blockIdx.x;
while(tid < N) {
c[tid] = a[tid] + b[tid];
tid += gridDim.x;
}
}