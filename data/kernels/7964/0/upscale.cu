#include "hip/hip_runtime.h"
#include "includes.h"



/*
* Description:
*/

__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_y, int scale_factor_x)
{
int x, y, t, z, w;


w = ii % d4;
ii = ii/d4;
z = ii % d3;
ii = ii/d3;
t = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
w = w/scale_factor_x;
z = z/scale_factor_y;
t = t/scale_factor_t;
d2 /= scale_factor_t;
d3 /= scale_factor_y;
d4 /= scale_factor_x;
return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}
__global__ void upscale(float *input, float *output, long no_elements, int scale_factor_t, int scale_factor_y, int scale_factor_x, int d1, int d2, int d3, int d4)
{
// output offset:
long ii = threadIdx.x + blockDim.x * blockIdx.x;
ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
if (ii >= no_elements) return;
int ipidx = translate_idx(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x);
output[ii]=input[ipidx];
}