#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(const int N, float *a, float *result) {

int thread = threadIdx.x;
int block  = blockIdx.x;
int blockSize = blockDim.x;
int gridSize = gridDim.x;


//unique global thread ID
int id = thread + block*blockSize;

__volatile__ __shared__ float s_sum[256];

float sum = 0;
for (int i=0; i<4; i++){
if(id+i*blockSize*gridSize<N){
sum += a[id+i*blockSize*gridSize]; //add the thread's id to start
}
}
s_sum[thread] = sum;

__syncthreads(); //make sure the write to shared is finished

if (thread<128) {//first half
s_sum[thread] += s_sum[thread+128];
}

__syncthreads(); //make sure the write to shared is finished


if (thread<64) {//next half
s_sum[thread] += s_sum[thread+64];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<32) {//next half
s_sum[thread] += s_sum[thread+32];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<16) {//next half
s_sum[thread] += s_sum[thread+16];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<8) {//next half
s_sum[thread] += s_sum[thread+8];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<4) {//next half
s_sum[thread] += s_sum[thread+4];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<2) {//next half
s_sum[thread] += s_sum[thread+2];
}

__syncthreads(); //make sure the write to shared is finished

if (thread<1) {//final piece
s_sum[thread] += s_sum[thread+1];
result[block] = s_sum[thread];
}
}