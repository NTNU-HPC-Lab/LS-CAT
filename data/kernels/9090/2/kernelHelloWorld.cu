#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelHelloWorld() {

int thread = threadIdx.x;//local thread number in a block
int block = blockIdx.x;//block number

printf("Hello World from thread %d of block %d!\n", thread, block);
}