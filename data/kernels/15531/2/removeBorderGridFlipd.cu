#include "hip/hip_runtime.h"
#include "includes.h"
/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose


#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
// 	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif


__global__ void removeBorderGridFlipd(float* dst, const unsigned int size, const unsigned int delta, const unsigned int blockSize, const float remplace){

int i = blockIdx.x * blockDim.x + threadIdx.x;
int blockId = blockIdx.y * blockDim.y + threadIdx.y;
int pos = i + blockId * delta;
if (pos < size && blockId < blockSize){
dst[pos]=remplace;
}
}