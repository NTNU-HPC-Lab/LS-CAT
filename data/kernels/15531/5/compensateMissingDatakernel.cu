#include "hip/hip_runtime.h"
#include "includes.h"
/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose


#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
// 	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif


__global__ void compensateMissingDatakernel(float* errosArray, float* crossErrosArray, const unsigned int size, float val){
int j = blockIdx.x*blockDim.x + threadIdx.x;
if (j < size)
{
errosArray[j]=-std::fabs(errosArray[j]/(crossErrosArray[j]*crossErrosArray[j]*crossErrosArray[j]*crossErrosArray[j]));
if(crossErrosArray[j]==0.0f) errosArray[j]=val;
}
}