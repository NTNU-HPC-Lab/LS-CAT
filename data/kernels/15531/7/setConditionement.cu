#include "hip/hip_runtime.h"
#include "includes.h"
/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose


#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
// 	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif


__global__ void setConditionement(unsigned size, unsigned* listIndex, float* listValueAtIndex, float* realSpaceArray, unsigned nbVar, unsigned var){
int j = blockIdx.x * blockDim.x + threadIdx.x;
if (j < size)
{
realSpaceArray[listIndex[j]]=listValueAtIndex[j*nbVar+var];
}
}