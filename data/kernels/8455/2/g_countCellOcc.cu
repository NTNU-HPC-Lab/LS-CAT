#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_countCellOcc(uint *_hash, uint *_cellOcc, uint _pixCount, uint _hashCellCount)
{
uint idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < _pixCount && _hash[idx] < _hashCellCount)
atomicAdd(&(_cellOcc[_hash[idx]]), 1);
}