#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match8small(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
#define NRX 2
__shared__ float4 buffer1[M7W*NDIM/4];
__shared__ float4 buffer2[M7H*NDIM/4];
int tx = threadIdx.x;
int ty = threadIdx.y;
int bp1 = M7W*blockIdx.x;
for (int d=tx;d<NDIM/4;d+=M7W)
for (int j=ty;j<M7W;j+=M7H/M7R/NRX)
buffer1[j*NDIM/4 + (d + j)%(NDIM/4)] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d];

float max_score[NRX];
int index[NRX];
for (int i=0;i<NRX;i++) {
max_score[i] = 0.0f;
index[i] = -1;
}
int idx = ty*M7W + tx;
int ix = idx%(M7W/NRX);
int iy = idx/(M7W/NRX);
for (int bp2=0;bp2<NPTS;bp2+=M7H) {
for (int d=tx;d<NDIM/4;d+=M7W)
for (int j=ty;j<M7H;j+=M7H/M7R/NRX)
buffer2[j*NDIM/4 + d] = ((float4*)d_pts2)[(bp2 + j)*(NDIM/4) + d];
__syncthreads();

float score[M7R][NRX];
for (int dy=0;dy<M7R;dy++)
for (int i=0;i<NRX;i++)
score[dy][i] = 0.0f;
for (int d=0;d<NDIM/4;d++) {
float4 v1[NRX];
for (int i=0;i<NRX;i++)
v1[i] = buffer1[((M7W/NRX)*i + ix)*NDIM/4 + (d + (M7W/NRX)*i + ix)%(NDIM/4)];
for (int dy=0;dy<M7R;dy++) {
float4 v2 = buffer2[(M7R*iy + dy)*(NDIM/4) + d];
for (int i=0;i<NRX;i++) {
score[dy][i] += v1[i].x*v2.x;
score[dy][i] += v1[i].y*v2.y;
score[dy][i] += v1[i].z*v2.z;
score[dy][i] += v1[i].w*v2.w;
}
}
}
for (int dy=0;dy<M7R;dy++) {
for (int i=0;i<NRX;i++) {
if (score[dy][i]>max_score[i]) {
max_score[i] = score[dy][i];
index[i] = bp2 + M7R*iy + dy;
}
}
}
__syncthreads();
}

float *scores = (float*)buffer1;
int *indices = (int*)&scores[M7W*M7H/M7R];
if (idx<M7W*M7H/M7R/NRX) {
for (int i=0;i<NRX;i++) {
scores[iy*M7W + (M7W/NRX)*i + ix] = max_score[i];
indices[iy*M7W + (M7W/NRX)*i + ix] = index[i];
}
}
__syncthreads();

if (ty==0) {
float max_score = scores[tx];
int index = indices[tx];
for (int y=0;y<M7H/M7R;y++)
if (scores[y*M7W + tx]>max_score) {
max_score = scores[y*M7W + tx];
index = indices[y*M7W + tx];
}
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}