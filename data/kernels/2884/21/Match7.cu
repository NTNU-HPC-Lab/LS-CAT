#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match7(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
__shared__ float4 buffer1[M7W*NDIM/4]; //%%%%
__shared__ float4 buffer2[M7H*NDIM/4];
int tx = threadIdx.x;
int ty = threadIdx.y;
int bp1 = M7W*blockIdx.x;
for (int d=tx;d<NDIM/4;d+=M7W)
for (int j=ty;j<M7W;j+=M7H/M7R)      //%%%%
buffer1[j*NDIM/4 + (d + j)%(NDIM/4)] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d];

float max_score = 0.0f;
int index = -1;
for (int bp2=0;bp2<NPTS;bp2+=M7H) {
for (int d=tx;d<NDIM/4;d+=M7W)
for (int j=ty;j<M7H;j+=M7H/M7R)       //%%%%
buffer2[j*NDIM/4 + d] = ((float4*)d_pts2)[(bp2 + j)*(NDIM/4) + d];
__syncthreads();

float score[M7R];
for (int dy=0;dy<M7R;dy++)
score[dy] = 0.0f;
for (int d=0;d<NDIM/4;d++) {
float4 v1 = buffer1[tx*NDIM/4 + (d + tx)%(NDIM/4)];
for (int dy=0;dy<M7R;dy++) {
float4 v2 = buffer2[(M7R*ty + dy)*(NDIM/4) + d];
score[dy] += v1.x*v2.x; score[dy] += v1.y*v2.y;
score[dy] += v1.z*v2.z; score[dy] += v1.w*v2.w;
}
}
for (int dy=0;dy<M7R;dy++) {
if (score[dy]>max_score) {
max_score = score[dy];
index = bp2 + M7R*ty + dy;
}
}
__syncthreads();
}

float *scores = (float*)buffer1;
int *indices = (int*)&scores[M7W*M7H/M7R];
scores[ty*M7W + tx] = max_score;
indices[ty*M7W + tx] = index;
__syncthreads();

if (ty==0) {
max_score = scores[tx];
index = indices[tx];
for (int y=0;y<M7H/M7R;y++)
if (scores[y*M7W + tx]>max_score) {
max_score = scores[y*M7W + tx];
index = indices[y*M7W + tx];
}
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}