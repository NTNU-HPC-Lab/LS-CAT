#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match2(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
__shared__ float buffer1[M2W*NDIM];  //%%%%
__shared__ float buffer2[M2H*NDIM];  //%%%%
__shared__ float scores[M2W*M2H];    //%%%%
int tx = threadIdx.x;
int ty = threadIdx.y;
int idx = tx + M2W*ty;
int bp1 = M2W*blockIdx.x;
if (ty<M2W)
for (int d=tx;d<NDIM;d+=M2W)
for (int j=ty;j<M2W;j+=M2H)
buffer1[j*NDIM + d] = d_pts1[(bp1 + j)*NDIM + d];   //%%%%
__syncthreads();

float max_score = 0.0f;
int index = -1;
for (int bp2=0;bp2<NPTS;bp2+=M2H) {
for (int d=tx;d<NDIM;d+=M2W)
buffer2[ty*NDIM + d] = d_pts2[(bp2 + ty)*NDIM + d]; //%%%%
__syncthreads();

float score = 0.0f;
for (int d=0;d<NDIM;d++)
score += buffer1[tx*NDIM + d]*buffer2[ty*NDIM + d];   //%%%%
scores[idx] = score;
__syncthreads();

if (ty==0) {
for (int i=0;i<M2H;i++) {
if (scores[i*M2W + tx]>max_score) {
max_score = scores[i*M2W + tx];
index = bp2 + i;
}
}
}
__syncthreads();
}

if (ty==0) {
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}