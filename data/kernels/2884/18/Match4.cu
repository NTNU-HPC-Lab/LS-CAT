#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match4(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
__shared__ float4 buffer1[M2W*(NDIM/4 + 1)];  //%%%%
__shared__ float4 buffer2[M2H*NDIM/4];        //%%%%
__shared__ float scores[M2W*M2H];
int tx = threadIdx.x;
int ty = threadIdx.y;
int idx = tx + M2W*ty;
int bp1 = M2W*blockIdx.x;
if (ty<M2W)
for (int d=tx;d<NDIM/4;d+=M2W)
for (int j=ty;j<M2W;j+=M2H)
buffer1[j*(NDIM/4 + 1) + d] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d]; //%%%%
__syncthreads();

float max_score = 0.0f;
int index = -1;
for (int bp2=0;bp2<NPTS;bp2+=M2H) {
for (int d=tx;d<NDIM/4;d+=M2W)
buffer2[ty*NDIM/4 + d] = ((float4*)d_pts2)[(bp2 + ty)*(NDIM/4) + d]; //%%%%
__syncthreads();

float score = 0.0f;
for (int d=0;d<NDIM/4;d++) {
float4 v1 = buffer1[tx*(NDIM/4 + 1) + d]; //%%%%
float4 v2 = buffer2[ty*(NDIM/4) + d];     //%%%%
score += v1.x*v2.x; score += v1.y*v2.y;
score += v1.z*v2.z; score += v1.w*v2.w;
}
scores[idx] = score;
__syncthreads();

if (ty==0) {
for (int i=0;i<M2H;i++) {
if (scores[i*M2W + tx]>max_score) {
max_score = scores[i*M2W + tx];
index = bp2 + i;
}
}
}
__syncthreads();
}

if (ty==0) {
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}