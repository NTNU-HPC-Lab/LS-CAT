#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match1(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
int p1 = threadIdx.x + M1W*blockIdx.x;
float max_score = 0.0f;
int index = -1;

for (int p2=0;p2<NPTS;p2++) {
float score = 0.0f;
for (int d=0;d<NDIM;d++)
score += d_pts1[p1*NDIM + d]*d_pts2[p2*NDIM + d];
if (score>max_score) {
max_score = score;
index = p2;
}
}

d_score[p1] = max_score;
d_index[p1] = index;
}