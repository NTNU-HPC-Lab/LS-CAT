#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match8blocked(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
#define NRX 2
#define NUM (NRX*M7R)                       // 32*8 threads
__shared__ float4 buffer1[M7W*NDIM/4];    // 32*32
__shared__ float4 buffer2[M7H*NUM];       // 32*8
int tx = threadIdx.x;
int ty = threadIdx.y;
int bp1 = M7W*blockIdx.x;
for (int d=tx;d<NDIM/4;d+=M7W)
for (int j=ty;j<M7W;j+=M7H/M7R)
buffer1[j*NDIM/4 + (d + j)%(NDIM/4)] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d];

float max_score[NRX];
int index[NRX];
for (int i=0;i<NRX;i++) {
max_score[i] = 0.0f;
index[i] = -1;
}
int idx = ty*M7W + tx;
int ix = idx%(M7W/NRX);
int iy = idx/(M7W/NRX);
for (int bp2=0;bp2<NPTS;bp2+=M7H) {
float score[M7R][NRX];
for (int dy=0;dy<M7R;dy++)
for (int i=0;i<NRX;i++)
score[dy][i] = 0.0f;

int d = (idx%NUM);
int j = (idx/NUM);
buffer2[j*NUM + d] = ((float4*)d_pts2)[(bp2 + j)*(NDIM/4) + d];
__syncthreads();
for (int dp=0;dp<NDIM/4;dp+=NUM) {
float4 temp;
if (dp<(NDIM/4-NUM))
temp = ((float4*)d_pts2)[(bp2 + j)*(NDIM/4) + dp + d + NUM];

if (idx<M7W*M7H/M7R/NRX) {
for (int d=0;d<NUM;d++) {
float4 v1[NRX];
#pragma unroll
for (int i=0;i<NRX;i++)
v1[i] = buffer1[(((M7W/NRX)*i + ix)<<5) + ((dp + d + (M7W/NRX)*i + ix)&31)];
//v1[i] = buffer1[((M7W/NRX)*i + ix)*NDIM/4 + (dp + d + (M7W/NRX)*i + ix)%(NDIM/4)];
#pragma unroll
for (int dy=0;dy<M7R;dy++) {
float4 v2 = buffer2[(M7R*iy + dy)*NUM + d];
#pragma unroll
for (int i=0;i<NRX;i++) {
score[dy][i] += v1[i].x*v2.x;
score[dy][i] += v1[i].y*v2.y;
score[dy][i] += v1[i].z*v2.z;
score[dy][i] += v1[i].w*v2.w;
}
}
}
}
__syncthreads();

if (dp<(NDIM/4-NUM)) {
buffer2[j*NUM + d] = temp;
__syncthreads();
}
}
for (int dy=0;dy<M7R;dy++) {
for (int i=0;i<NRX;i++) {
if (score[dy][i]>max_score[i]) {
max_score[i] = score[dy][i];
index[i] = bp2 + M7R*iy + dy;
}
}
}
__syncthreads();
}

float *scores = (float*)buffer1;
int *indices = (int*)&scores[M7W*M7H/M7R];
if (idx<M7W*M7H/M7R/NRX) {
for (int i=0;i<NRX;i++) {
scores[iy*M7W + (M7W/NRX)*i + ix] = max_score[i];
indices[iy*M7W + (M7W/NRX)*i + ix] = index[i];
}
}
__syncthreads();

if (ty==0) {
float max_score = scores[tx];
int index = indices[tx];
for (int y=0;y<M7H/M7R;y++)
if (scores[y*M7W + tx]>max_score) {
max_score = scores[y*M7W + tx];
index = indices[y*M7W + tx];
}
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}