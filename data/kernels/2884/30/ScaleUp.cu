#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
const int tx = threadIdx.x;
const int ty = threadIdx.y;
int x = blockIdx.x*SCALEUP_W + 2*tx;
int y = blockIdx.y*SCALEUP_H + 2*ty;
if (x<2*width && y<2*height) {
int xl = blockIdx.x*(SCALEUP_W/2) + tx;
int yu = blockIdx.y*(SCALEUP_H/2) + ty;
int xr = min(xl + 1, width - 1);
int yd = min(yu + 1, height - 1);
float vul = d_Data[yu*pitch + xl];
float vur = d_Data[yu*pitch + xr];
float vdl = d_Data[yd*pitch + xl];
float vdr = d_Data[yd*pitch + xr];
d_Result[(y + 0)*newpitch + x + 0] = vul;
d_Result[(y + 0)*newpitch + x + 1] = 0.50f*(vul + vur);
d_Result[(y + 1)*newpitch + x + 0] = 0.50f*(vul + vdl);
d_Result[(y + 1)*newpitch + x + 1] = 0.25f*(vul + vur + vdl + vdr);
}
}