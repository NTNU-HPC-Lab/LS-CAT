#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Match6(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
__shared__ float4 buffer1[M5W*(NDIM/4 + 1)];
__shared__ float4 buffer2[M5H*NDIM/4];
int tx = threadIdx.x;
int ty = threadIdx.y;
int bp1 = M5W*blockIdx.x;
if (ty<M5W)
for (int d=tx;d<NDIM/4;d+=M5W)
for (int j=ty;j<M5W;j+=M5H)
buffer1[j*(NDIM/4 + 1) + d] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d];

float max_score = 0.0f;
int index = -1;
for (int bp2=0;bp2<NPTS;bp2+=M5H) {
for (int d=tx;d<NDIM/4;d+=M5W)
buffer2[ty*NDIM/4 + d] = ((float4*)d_pts2)[(bp2 + ty)*(NDIM/4) + d];
__syncthreads();

if (ty<M5H/M5R) {
float score[M5R];
for (int dy=0;dy<M5R;dy++)
score[dy] = 0.0f;
for (int d=0;d<NDIM/4;d++) {
float4 v1 = buffer1[tx*(NDIM/4 + 1) + d];
for (int dy=0;dy<M5R;dy++) {
float4 v2 = buffer2[(M5R*ty + dy)*(NDIM/4) + d];
score[dy] += v1.x*v2.x; score[dy] += v1.y*v2.y;
score[dy] += v1.z*v2.z; score[dy] += v1.w*v2.w;
}
}
for (int dy=0;dy<M5R;dy++) {
if (score[dy]>max_score) {   //%%%%
max_score = score[dy];
index = bp2 + M5R*ty + dy;
}
}
}
__syncthreads();
}

float *scores = (float*)buffer1;
int *indices = (int*)&scores[M5W*M5H/M5R];
if (ty<M5H/M5R) {
scores[ty*M5W + tx] = max_score;  //%%%%
indices[ty*M5W + tx] = index;     //%%%%
}
__syncthreads();

if (ty==0) {
max_score = scores[tx];
index = indices[tx];
for (int y=0;y<M5H/M5R;y++)
if (scores[y*M5W + tx]>max_score) {
max_score = scores[y*M5W + tx]; //%%%%
index = indices[y*M5W + tx];    //%%%%
}
d_score[bp1 + tx] = max_score;
d_index[bp1 + tx] = index;
}
}