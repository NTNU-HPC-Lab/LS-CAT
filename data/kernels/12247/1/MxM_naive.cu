#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MxM_naive(double* A, double* B, double* C, const int N) {

int i = blockIdx.y * blockDim.y + threadIdx.y;    // Row i of matrix C
int j = blockIdx.x * blockDim.x + threadIdx.x;    // Column j of matrix C

double C_temp = 0;
for (int k=0; k<N; k++) {
// use 1D indexing
C_temp += A[i*N + k] * B[k*N + j];
}

// write back to global memory
// no synchronization needed here because one thread handles one element
C[i*N + j] = C_temp;

}