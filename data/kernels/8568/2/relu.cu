#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu(float *inout, float *bias, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
inout[i * cols + j] = fmaxf(0.0, inout[i * cols + j] + bias[i]);
}