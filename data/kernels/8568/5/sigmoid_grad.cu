#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid_grad(float *pre_grad, float *output, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
float t = output[i * cols + j];
pre_grad[i * cols + j] *= t * (1 - t);
}