#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanh(float *inout, float *bias, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
inout[i * cols + j] = tanhf(inout[i * cols + j]) + bias[i];
}