#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_grad(float *pre_grad, float *output, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;

if (j >= cols || i >= rows) return;
if (output[i * cols + j] <= 0)
pre_grad[i * cols + j] = 0;
}