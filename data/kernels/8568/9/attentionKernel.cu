#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void attentionKernel(float *x, int rows, int cols) {
int j = blockIdx.x * blockDim.x + threadIdx.x;

if (j >= cols) return;
float sum = 0;
for (int k = 0; k < rows; k++) {
sum += x[k * cols + j];
}
for (int k = 0; k < rows; k++) {
x[k * cols + j] *= sum;
}
}