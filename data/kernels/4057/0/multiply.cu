#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_SIZE 16

/*
* prints matrices
* Because matrices filled with dummy 0s function takes 3 dim arguments:
*      actual x and y dimension and dim as big square matrix's dimension
*/
__global__ void multiply(float *left, float *right, float *res, int dim) {

int i,j;
float temp = 0;

__shared__ float Left_shared_t [BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

// Row i of matrix left
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;


for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

// Column j of matrix left
j = tileNUM * BLOCK_SIZE + threadIdx.x;
i = tileNUM * BLOCK_SIZE + threadIdx.y;
// Load left[i][j] to shared mem

Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];// Coalesced access
// Load right[i][j] to shared mem

Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col]; // Coalesced access
// Synchronize before computation
__syncthreads();

// Accumulate one tile of res from tiles of left and right in shared mem
for (int k = 0; k < BLOCK_SIZE; k++) {

temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x]; //no shared memory bank conflict
}
// Synchronize
__syncthreads();
}
// Store accumulated value to res
res[row * dim + col] = temp;
}