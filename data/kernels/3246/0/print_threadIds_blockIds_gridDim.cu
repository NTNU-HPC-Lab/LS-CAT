#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void print_threadIds_blockIds_gridDim()
{
printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d,\
blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d,\
gridDim.x: %d, gridDim.y: %d, gridDim.z: %d \n",
threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z,
gridDim.x, gridDim.y, gridDim.z);
}