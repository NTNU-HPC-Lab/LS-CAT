#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modcpy(void *destination, void *source, size_t destination_size, size_t source_size){

int idx = blockIdx.x * blockDim.x + threadIdx.x;
int pos;

int ds = destination_size/sizeof(int4), ss = source_size/sizeof(int4);
for(int i = idx; i < ds; i += gridDim.x * blockDim.x){
pos = i % ss;
reinterpret_cast<int4*>(destination)[i] = reinterpret_cast<int4*>(source)[pos];
}
}