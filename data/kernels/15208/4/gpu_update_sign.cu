#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_update_sign(int *G, double *w ,int *neighbors , int k , int n ,int *temp, int *flag,int it_b ,int it_t)
{
int result;
double sum = 0.0;
int buf=0;
//Find the indexes
int x = blockIdx.x+it_b*gridDim.x;
int y = threadIdx.x+it_t*blockDim.x;

if (blockIdx.x+it_b*gridDim.x<n && threadIdx.x+it_t*blockDim.x<n)
{
//Calculate result
for (int i = 0; i < k; i++){
for (int j = 0; j < k; j++){
sum += ((double)G[neighbors[x*n*k*k+y*k*k+i*k+j]])*w[i*k+j];
}
}
//Evaluate and write back
if ( sum > 1e-6){
result = 1;
if (result != G[neighbors[x*n*k*k+y*k*k+12]])
buf++;
}
else if( sum < -(1e-6)){
result = -1;
if (result != G[neighbors[x*n*k*k+y*k*k+12]])
buf++;
}
else
result = G[neighbors[x*n*k*k+y*k*k+12]];

*flag+=buf;
temp[x*n+y] =result;
}
}