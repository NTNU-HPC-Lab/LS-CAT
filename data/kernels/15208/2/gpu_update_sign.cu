#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_update_sign(int *G, double *w ,int *neighbors , int k , int n ,int *temp, int *flag,int it_b ,int it_t)
{

int buf=0;

for (int off1 = 0; off1 < it_b; off1++)
{
for(int off2 = 0; off2<it_t;off2++){
int result;
double sum = 0.0;

int x = blockIdx.x+off1*gridDim.x;
int y = threadIdx.x+off2*blockDim.x;

if(x<n && y<n){
for (int i = 0; i < k; i++){
for (int j = 0; j < k; j++){
sum += ((double)G[neighbors[x*n*k*k+y*k*k+i*k+j]])*w[i*k+j];
}
}

if ( sum > 1e-6){
result = 1;
if (result != G[neighbors[x*n*k*k+y*k*k+12]])
buf++;
}
else if( sum < -(1e-6)){
result = -1;
if (result != G[neighbors[x*n*k*k+y*k*k+12]])
buf++;
}
else{
result = G[neighbors[x*n*k*k+y*k*k+12]];
}
temp[x*n+y] =result;
}
}
}
*flag+=buf;
__syncthreads();
}