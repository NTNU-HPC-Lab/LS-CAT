#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_get_neighors(int *neighbors, int n , int k)
{
for (int off1 = 0; off1 < n/gridDim.x+1 ; off1++)
{
for(int off2 = 0; off2 < n/blockDim.x+1 ;off2++){

int m = blockIdx.x+off1*gridDim.x;
int l = threadIdx.x+off2*blockDim.x;

int counter_i =0;
if(m<n && l<n){
for (int i = m-(k/2); i <= m+(k/2); i++)
{
int counter_j=0;
for (int j = l-(k/2); j <= l+(k/2); j++)
{
int index , index_i , index_j;
index = m*n*k*k + l*k*k + counter_i*k +counter_j;
index_i =(n+i)%n;
index_j=(n+j)%n;
neighbors[index] = index_i*n+index_j;
counter_j++;
}
counter_i++;
}
}
}
}
}