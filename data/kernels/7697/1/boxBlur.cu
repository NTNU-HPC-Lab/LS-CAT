#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void boxBlur(uchar4 *d_img, uchar4 *d_blurred, int numRows, int numCols, int k){


printf("Hilo: %d\n", threadIdx.x);
printf("KERNEL PAANIC R: %d, G: %d, B: %d\n", d_img[numCols*69+45].x, d_img[numCols*1 +7].y, d_img[numCols*1 +7].z);

int id = threadIdx.x;
int accumulator, cornerX, cornerY, sumRGB[] = {0,0,0};


for(int x=id; x<numRows; x+=blockDim.x){
//printf("hilo: %d, col: %d, blockDim.x: %d\n", id, x, blockDim.x);

for(int y=0; y<numCols; y++){
//printf("hilo: %d, row: %d, col: %d, blockDim.x: %d\n", id, x, y, blockDim.x);
//printf("(%d,%d,%d) R: %d, G: %d, B: %d\n", id, x, y, d_img[numCols*x+y].x, d_img[numCols*x+y].y, d_img[numCols*x+y].z);
//cout << y << endl;

int pixel = numCols * x + y;
uchar4 pixelColor = d_img[pixel];
//printf("R: %d, G: %d, B: %d\n", pixelColor.x, pixelColor.y, pixelColor.z);
sumRGB[0]=0;
sumRGB[1]=0;
sumRGB[2]=0;
accumulator=0;
if(k%2==0){
cornerX = x-(k-1)/2;
cornerY = y-(k-1)/2;
}else{
cornerX = x-(k-2)/2;
cornerY = y-(k-2)/2;
}

for(int i = 0; i < k; i++){
for(int j = 0; j < k; j++){
if(cornerX+i>=0 && cornerX+i<numRows && cornerY+j>=0 && cornerY+j<numCols) {
int index = numCols * (cornerX+i) + (cornerY+j);
uchar4 color = d_img[index];
sumRGB[0] += (int) color.x;
sumRGB[1] += (int) color.y;
sumRGB[2] += (int) color.z;
accumulator++;
}
}
}



if(accumulator == 0){
printf("WHAAAA?\n" );
sumRGB[0] = pixelColor.x;
sumRGB[1] = pixelColor.y;
sumRGB[2] = pixelColor.z;
}else{
sumRGB[0] = sumRGB[0] / accumulator;
sumRGB[1] = sumRGB[1] / accumulator;
sumRGB[2] = sumRGB[2] / accumulator;
}

uchar4 color;
color.x = sumRGB[0];
color.y = sumRGB[1];
color.z = sumRGB[2];

//myLock.lock();
d_blurred[pixel] = color;
//myLock.unlock();
//printf("Original R: %d, G: %d, B: %d\n", pixelColor.x, pixelColor.y, pixelColor.z);
//printf("Blurred  R: %d, G: %d, B: %d\n", d_blurred[pixel].x, d_blurred[pixel].y, d_blurred[pixel].z);

}

}


}