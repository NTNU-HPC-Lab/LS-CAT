#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square( int *d_num_steps, unsigned long long *d_fact, double *d_out){
int idx = threadIdx.x;
int num_steps = *d_num_steps;
for(int k=idx+1; k< num_steps; k+=blockDim.x){
d_out[idx] += (double) k*0.5/ (double) d_fact[k-1];
}

}