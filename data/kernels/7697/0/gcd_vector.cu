#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
int threads;


__global__ void gcd_vector(int * d_out, int integer_m){
int idx = threadIdx.x;
for(int i = idx; i<integer_m; i+=blockDim.x){
int u = i, v = integer_m;
while ( v != 0) {
int r = u % v;
u = v;
v = r;
}
if(u == 1){
d_out[idx]++;
}
}
}