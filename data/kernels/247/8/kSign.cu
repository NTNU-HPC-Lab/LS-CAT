#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSign(float* gData, float* target, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
target[i] = (gData[i] > 0) - (gData[i] < 0);
}