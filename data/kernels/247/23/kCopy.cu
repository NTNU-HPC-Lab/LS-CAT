#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCopy(float* srcStart, float* destStart, const int copyWidth, const int srcJumpWidth, const int destJumpWidth, const int numElements) {
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
for (int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
destStart[(i / copyWidth) * destJumpWidth + i % copyWidth] = srcStart[(i / copyWidth) * srcJumpWidth + i % copyWidth];
}
}