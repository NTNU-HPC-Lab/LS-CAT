#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k_copy_reshape_rowmajor(unsigned int numEls, unsigned int a_nd, const float * a_data, const int * a_dim, const int * a_str, unsigned int z_nd, float * z_data, const int * z_dim, const int * z_str)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < numEls; i += numThreads)
{
const float * a_i = a_data;
int a_ii = i;
for (unsigned int _d = 0; _d < a_nd; ++_d) //make the rightmost coords change fastest
{
unsigned int d = a_nd - _d-1;
int a_i_d = a_ii % a_dim[d];
a_ii = a_ii / a_dim[d];
a_i += a_i_d * a_str[d];
}
int z_ii = i;
float * z_i = z_data;
for (unsigned int _d = 0; _d < z_nd; ++_d) //make the rightmost coords change fastest
{
unsigned int d = z_nd - _d-1;
//i tried to make the for loop count down, but it didn't work!?
int z_i_d = z_ii % z_dim[d];
z_i += z_i_d * z_str[d];
z_ii = z_ii / z_dim[d];
}
z_i[0] = a_i[0]; //copy one lousy float!
}
}