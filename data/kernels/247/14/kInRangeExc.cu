#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kInRangeExc(float* gData, float lower, float upper, float* target, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
target[i] = gData[i] > lower && gData[i] < upper;
}