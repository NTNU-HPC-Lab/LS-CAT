#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBiggerThan(float* gMat1, float* gMat2, float* gMatTarget, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < numElements)
gMatTarget[idx] = gMat1[idx] > gMat2[idx];
}