#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuMatMul(float *a, float *b, float *c, int m, int n, int p) {
uint i = blockDim.x * blockIdx.x + threadIdx.x;
uint j = blockDim.y * blockIdx.y + threadIdx.y;
if (i < m && j < p) {
float val = 0;
for (int k = 0; k < n; ++k) {
val += a[i * n + k] * b[k * p + j];
}
c[i * p + j] = val;
}
}