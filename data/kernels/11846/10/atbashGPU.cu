#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atbashGPU(char const *in, char *out, int n) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < n) {
out[n - 1 - i] = in[i];
}
}