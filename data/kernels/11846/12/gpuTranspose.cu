#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuTranspose(float *a, float *b, int m, int n) {
uint i = blockDim.x * blockIdx.x + threadIdx.x;
uint j = blockDim.y * blockIdx.y + threadIdx.y;
if (i < m && j < n) {
b[j * m + i] = a[i * n + j];
}
}