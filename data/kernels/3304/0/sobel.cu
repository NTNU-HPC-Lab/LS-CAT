#include "hip/hip_runtime.h"
#include "includes.h"
// Jim Samson
// CSF441 Computer Architecture
// Assignment 4
// Most code is written by Dr. Mock
// This HW Assignment uses cuda and the Sobel filter to convert an image.

/***********************************************************************
* sobel-cpu.cu
*
* Implements a Sobel filter on the image that is hard-coded in main.
* You might add the image name as a command line option if you were
* to use this more than as a one-off assignment.
*
* See https://stackoverflow.com/questions/17815687/image-processing-implementing-sobel-filter
* or https://blog.saush.com/2011/04/20/edge-detection-with-the-sobel-operator-in-ruby/
* for info on how the filter is implemented.
*
* Compile/run with:  nvcc sobel-cpu.cu -lfreeimage
*
***********************************************************************/


#define threadsPerBlock 22
// Returns the index into the 1d pixel array
// Given te desired x,y, and image width
__device__ int pixelIndex(int x, int y, int width)
{
return (y*width + x);
}
__global__ void sobel(char *returnPixels, int width, char *pixels)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int x00 = -1;  int x20 = 1;
int x01 = -2;  int x21 = 2;
int x02 = -1;  int x22 = 1;
x00 *= pixels[pixelIndex(x-1,y-1,width)];
x01 *= pixels[pixelIndex(x-1,y,width)];
x02 *= pixels[pixelIndex(x-1,y+1,width)];
x20 *= pixels[pixelIndex(x+1,y-1,width)];
x21 *= pixels[pixelIndex(x+1,y,width)];
x22 *= pixels[pixelIndex(x+1,y+1,width)];

int y00 = -1;  int y10 = -2;  int y20 = -1;
int y02 = 1;  int y12 = 2;  int y22 = 1;
y00 *= pixels[pixelIndex(x-1,y-1,width)];
y10 *= pixels[pixelIndex(x,y-1,width)];
y20 *= pixels[pixelIndex(x+1,y-1,width)];
y02 *= pixels[pixelIndex(x-1,y+1,width)];
y12 *= pixels[pixelIndex(x,y+1,width)];
y22 *= pixels[pixelIndex(x+1,y+1,width)];

int px = x00 + x01 + x02 + x20 + x21 + x22;
int py = y00 + y10 + y20 + y02 + y12 + y22;
returnPixels[pixelIndex(x,y,width)] = sqrt(float(px*px + py*py));

}