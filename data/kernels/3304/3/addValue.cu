#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addValue(int * array_val, int*b_array_val) {
int x = threadIdx.x;
int sum = 0;

for(unsigned int i = 0; i < ROWS; i++) {
sum += array_val[i*COLUMNS+x];
}
b_array_val[x] = sum;
}