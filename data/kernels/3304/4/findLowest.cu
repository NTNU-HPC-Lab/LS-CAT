#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findLowest(int numMin, int *array_val, int *cudaResult ) {
int low = threadIdx.x * numMin;
int high = low + numMin -1;
int min = array_val[low];
for (unsigned int i = low; i < high; i++){
if(array_val[i] < min){
min = array_val[i];
}
}
cudaResult[threadIdx.x] = min;
printf("Thread %d returned: %d \n", threadIdx.x, min);
}