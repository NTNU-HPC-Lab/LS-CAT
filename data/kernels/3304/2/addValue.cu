#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addValue(int *array_val, int *b_array_val) {
int cacheIndex = threadIdx.x;
int i = blockDim.x/2;
while (i > 0) {
if (cacheIndex < i) {
array_val[blockIdx.x * COLUMNS +cacheIndex] += array_val[blockIdx.x * COLUMNS + cacheIndex +i];
}
__syncthreads();
i /=2;
}
if (cacheIndex == 0)
b_array_val[blockIdx.x] = array_val[blockIdx.x * COLUMNS];

}