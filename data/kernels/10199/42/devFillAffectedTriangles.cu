#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devFillAffectedTriangles(int nFlip, int *pTaff, int *pTaffEdge, int *pEnd, int2 *pEt)
{
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

while (i < nFlip) {
int e = pEnd[i];

pTaffEdge[i] = i;
pTaffEdge[i + nFlip] = i;

pTaff[i]         = pEt[e].x;
pTaff[i + nFlip] = pEt[e].y;

i += gridDim.x*blockDim.x;
}
}