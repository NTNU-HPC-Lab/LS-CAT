#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devInverseReindex(int N, int *destArray, int *srcArray, unsigned int *reindex, int realSize, int nDims, int maxValue, bool ignoreValue)
{
for (unsigned int n = 0; n < nDims; n++) {
int i = blockIdx.x*blockDim.x + threadIdx.x;

while (i < N) {
int ret = -1;
int tmp = srcArray[i + n*realSize];
if (tmp != -1 || ignoreValue == false) {
int addValue = 0;
while (tmp >= maxValue) {
tmp -= maxValue;
addValue += maxValue;
}
while (tmp < 0) {
tmp += maxValue;
addValue -= maxValue;
}
ret = (int) reindex[tmp] + addValue;
}
destArray[i + n*realSize] = ret;

i += gridDim.x*blockDim.x;
}
}
}