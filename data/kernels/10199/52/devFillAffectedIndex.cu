#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devFillAffectedIndex(int nRemove, int maxTriPerVert, int *pTriangleAffectedIndex)
{
int n = blockIdx.x*blockDim.x + threadIdx.x;

while (n < nRemove) {
for (int i = 0; i < maxTriPerVert; i++) {
pTriangleAffectedIndex[i + n*maxTriPerVert] = n;
pTriangleAffectedIndex[i + n*maxTriPerVert + nRemove*maxTriPerVert] = n;
}

n += blockDim.x*gridDim.x;
}
}