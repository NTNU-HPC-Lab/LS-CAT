#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devInverseReindexInt2Bool(int N, int2 *destArray, int2 *srcArray, unsigned int *reindex, int realSize, int nDims, int maxValue, bool ignoreValue)
{
for (unsigned int n = 0; n < nDims; n++) {
int i = blockIdx.x*blockDim.x + threadIdx.x;

while (i < N) {
int ret = -1;
int tmp = srcArray[i + n*realSize].x;
if (tmp != -1 || ignoreValue == false) {
int addValue = 0;
while (tmp >= maxValue) {
tmp -= maxValue;
addValue += maxValue;
}
while (tmp < 0) {
tmp += maxValue;
addValue -= maxValue;
}
ret = (int) reindex[tmp] + addValue;
}
destArray[i + n*realSize].x = ret;

ret = -1;
tmp = srcArray[i + n*realSize].y;
if (tmp != -1 || ignoreValue == false) {
int addValue = 0;
while (tmp >= maxValue) {
tmp -= maxValue;
addValue += maxValue;
}
while (tmp < 0) {
tmp += maxValue;
addValue -= maxValue;
}
ret = (int) reindex[tmp] + addValue;
}
destArray[i + n*realSize].y = ret;

i += gridDim.x*blockDim.x;
}
}
}