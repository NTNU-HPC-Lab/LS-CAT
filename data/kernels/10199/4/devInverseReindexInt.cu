#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devInverseReindexInt(int N, int *destArray, int *srcArray, int *reindex, int realSize, int nDims)
{
for (unsigned int n = 0; n < nDims; n++) {
int i = blockIdx.x*blockDim.x + threadIdx.x;

while (i < N) {
int tmp = srcArray[i + n*realSize];
destArray[i + n*realSize] = reindex[tmp];

i += gridDim.x*blockDim.x;
}
}
}