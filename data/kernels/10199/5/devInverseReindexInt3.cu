#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devInverseReindexInt3(int N, int3 *destArray, int3 *srcArray, int *reindex, int realSize, int nDims)
{
for (unsigned int n = 0; n < nDims; n++) {
int i = blockIdx.x*blockDim.x + threadIdx.x;

while (i < N) {
int tmp = srcArray[i + n*realSize].x;
destArray[i + n*realSize].x = reindex[tmp];
tmp = srcArray[i + n*realSize].y;
destArray[i + n*realSize].y = reindex[tmp];
tmp = srcArray[i + n*realSize].z;
destArray[i + n*realSize].z = reindex[tmp];

i += gridDim.x*blockDim.x;
}
}
}