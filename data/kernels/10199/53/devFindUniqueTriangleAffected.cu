#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devFindUniqueTriangleAffected(int maxIndex, int *pTriangleAffected, int *pTriangleAffectedIndex, int *pUniqueFlag)
{
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x + 1;

while (i < maxIndex) {
if (pTriangleAffected[i-1] == pTriangleAffected[i] &&
pTriangleAffected[i] != -1) {
int j = pTriangleAffectedIndex[i];
pUniqueFlag[j] = 0;
}

i += gridDim.x*blockDim.x;
}
}