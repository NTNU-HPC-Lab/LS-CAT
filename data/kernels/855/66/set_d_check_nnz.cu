#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_d_check_nnz(int *d_check_nnz, int *d_nnz_num, int pad_M, int SIGMA, int sigma_block_row)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= pad_M) {
return;
}

int a = 1;
if (d_nnz_num[blockIdx.y * pad_M + i] > 0) {
atomicAdd(&(d_check_nnz[blockIdx.y * sigma_block_row + i / SIGMA]), a);
}
}