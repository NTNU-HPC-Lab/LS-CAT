#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compress_write_permutation(int *d_write_permutation, int *d_full_write_permutation, int *d_gcs, int total_pad_row_num, int chunk)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= total_pad_row_num) {
return;
}

int chunk_id = i / chunk;
if (d_gcs[chunk_id + 1] - d_gcs[chunk_id] > 0) {
int tid = i % chunk;
d_write_permutation[d_gcs[chunk_id] * chunk + tid] = d_full_write_permutation[i];
}
}