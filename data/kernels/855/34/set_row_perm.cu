#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_row_perm(int *d_bin_size, int *d_bin_offset, int *d_max_row_nz, int *d_row_perm, int M, int min, int mmin)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i >= M) {
return;
}

int nz_per_row = d_max_row_nz[i];
int dest;

int j = 0;
for (j = 0; j < BIN_NUM - 2; j++) {
if (nz_per_row <= (min << j)) {
if (nz_per_row <= mmin) {
dest = atomicAdd(d_bin_size + j, 1);
d_row_perm[d_bin_offset[j] + dest] = i;
}
else {
dest = atomicAdd(d_bin_size + j + 1, 1);
d_row_perm[d_bin_offset[j + 1] + dest] = i;
}
return;
}
}
dest = atomicAdd(d_bin_size + BIN_NUM - 1, 1);
d_row_perm[d_bin_offset[BIN_NUM - 1] + dest] = i;

}