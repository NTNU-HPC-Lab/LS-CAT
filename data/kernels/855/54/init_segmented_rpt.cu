#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_segmented_rpt(int *d_nnz_num, int *d_seg_rpt, int total_pad_row_num)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i > total_pad_row_num) {
return;
}

if (i == 0) {
d_seg_rpt[i] = 0;
}

else {
d_seg_rpt[i] = d_nnz_num[i - 1];
}
}