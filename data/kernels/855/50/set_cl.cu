#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_cl(int *nnz_num, int *cl, int chunk, int pad_M)
{
int c_size = pad_M / chunk;
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= c_size) {
return;
}
int offset = chunk * i;
int max = 0;
int j, length;
for (j = 0; j < chunk; j++) {
length = nnz_num[offset + j];
if (length > max) {
max = length;
}
}
cl[i] = max;
}