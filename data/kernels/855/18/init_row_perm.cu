#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_row_perm(int *d_permutation, int M)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= M) {
return;
}

d_permutation[i] = i;
}