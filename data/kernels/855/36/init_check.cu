#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_check(int *d_check, int nz)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= nz) {
return;
}
d_check[i] = -1;
}