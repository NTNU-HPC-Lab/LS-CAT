#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_segmented_nnz_num(int *d_rpt, int *d_col, int *d_nnz_num, int *d_group_seg, int *d_offset, size_t seg_size, size_t seg_num, int M, int pad_M, int group_num_col)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= M) {
return;
}

int width = d_rpt[i + 1] - d_rpt[i];

int g, j;
int col;

int offset = d_rpt[i];
int index;

for (j = 0; j < width; j++) {
index = offset + j;
col = d_col[index];
g = col / seg_size;
d_offset[index] = d_nnz_num[g * pad_M + i];
d_nnz_num[g * pad_M + i]++;
d_group_seg[index] = g;
}
}