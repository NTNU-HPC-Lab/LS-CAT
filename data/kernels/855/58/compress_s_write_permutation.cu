#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compress_s_write_permutation(unsigned short *d_s_write_permutation, unsigned short *d_s_write_permutation_offset, int *d_write_permutation, int c_size, int chunk)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= c_size * chunk) {
return;
}

int chunk_id = i / chunk;
d_s_write_permutation[i] = (unsigned short)(d_write_permutation[i] % USHORT_MAX);
if (i % chunk == 0) {
d_s_write_permutation_offset[chunk_id] = (unsigned short)(d_write_permutation[i] / USHORT_MAX);
}
}