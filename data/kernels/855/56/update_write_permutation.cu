#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_write_permutation(int *write_permutation, int *nnz_num, int total_pad_row_num, int pad_M)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= total_pad_row_num) {
return;
}

write_permutation[i] -= (i / pad_M) * pad_M;
}