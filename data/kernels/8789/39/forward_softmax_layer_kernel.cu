#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(b >= batch) return;

int i;
float sum = 0;
float largest = -INFINITY;
for(i = 0; i < n; ++i){
int val = input[i+b*n];
largest = (val>largest) ? val : largest;
}
for(i = 0; i < n; ++i){
sum += exp(input[i+b*n]/temp-largest/temp);
}
sum = (sum != 0) ? largest/temp+log(sum) : largest-100;
for(i = 0; i < n; ++i){
output[i+b*n] = exp(input[i+b*n]/temp-sum);
}
}