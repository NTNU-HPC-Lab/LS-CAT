#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float machine_eps_flt() {
typedef union {
int i32;
float f32;
} flt_32;

flt_32 s;

s.f32 = 1.;
s.i32++;
return (s.f32 - 1.);
}
__device__ double machine_eps_dbl() {
typedef union {
long long i64;
double d64;
} dbl_64;

dbl_64 s;

s.d64 = 1.;
s.i64++;
return (s.d64 - 1.);
}
__global__ void calc_consts(float *fvals, double *dvals) {

int i = threadIdx.x + blockIdx.x*blockDim.x;
if (i==0) {
fvals[EPS] = machine_eps_flt();
dvals[EPS]= machine_eps_dbl();

float xf, oldxf;
double xd, oldxd;

xf = 2.; oldxf = 1.;
xd = 2.; oldxd = 1.;

/* double until overflow */
/* Note that real fmax is somewhere between xf and oldxf */
while (!isinf(xf))  {
oldxf *= 2.;
xf *= 2.;
}

while (!isinf(xd))  {
oldxd *= 2.;
xd *= 2.;
}

dvals[MAX] = oldxd;
fvals[MAX] = oldxf;

/* half until overflow */
/* Note that real fmin is somewhere between xf and oldxf */
xf = 1.; oldxf = 2.;
xd = 1.; oldxd = 2.;

while (xf != 0.)  {
oldxf /= 2.;
xf /= 2.;
}

while (xd != 0.)  {
oldxd /= 2.;
xd /= 2.;
}

dvals[MIN] = oldxd;
fvals[MIN] = oldxf;

}
return;
}