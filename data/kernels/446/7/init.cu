#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(unsigned int seed, hiprandState_t* states) {

/* we have to initialize the state */
hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
threadIdx.x, /* the sequence number should be different for each core (unless you want all
cores to get the same sequence of numbers for some reason - use thread id! */
0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&states[threadIdx.x]);
}