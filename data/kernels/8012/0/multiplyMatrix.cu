#include "hip/hip_runtime.h"
#include "includes.h"

#define N 1024 //wielkoæ obliczanych wektorów
#define imin(a, b) (a<b?a:b)
const int threadsPerBlock = 256; //iloæ w¹tków na k¹zdy blok
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);//iloæ wykorzystywanych bloków





__global__ void multiplyMatrix(float *a, float *b, float *c) {
__shared__ float cache[threadsPerBlock]; //Zmienna dzielona ze wszystkimi w¹tkami w tym bloku. Nie dzieli siê z innymi blokami!

int tid = threadIdx.x + blockIdx.x * blockDim.x; //id w¹tku który to wykonuje, id w¹tku + id bloku * pojemnoæ bolku
int cacheIndex = threadIdx.x; //id cache, które jest takie samo jak id obecnego w¹tku

float temp = 0;
while (tid < N) {
temp = a[tid] * b[tid]; //zapis mno¿enia w zmiennej
tid += blockDim.x * gridDim.x; //przesuwanie o iloæ wszystkich w¹tków w ca³ej siatce, nie trzeba ogarniaæ na czwórkê
}

cache[cacheIndex] = temp; //przypisanie wyniku mno¿enia do wspó³dzielonej tablicy cache

__syncthreads(); //czekanie a¿ wszystkie w¹tki dotr¹ to tego miejsca

//tu trochê w powalony sposób sumuj¹ siê wszystkie wyniki
int i = blockDim.x / 2;
while (i != 0) {
if (cacheIndex < i) {
cache[cacheIndex] += cache[cacheIndex + i];
}

__syncthreads();
i /= 2;
}
//przypisanie sumy wszystkich wyników mno¿enia do tablicy c
if (cacheIndex == 0)
c[blockIdx.x] = cache[0]; //jako, ¿e cache nie jest wspó³dzielony pomiêdzy blokami to wyników bêdzie tyle ile by³o wykorzystanych bloków, póniej to siê sumuje na cpu

}