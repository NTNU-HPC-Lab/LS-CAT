#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_BlockSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputSize, int outputSize, int nInputBlock, int nOutputBlock, int inputWindowSize, int outputWindowSize, float scale)
{
__shared__ float buffer[BLOCKSPARSE_THREADS];
__shared__ float gradOutputBuffer[BLOCKSPARSE_MAXOUTPUTBLOCKSIZE];
int tx = threadIdx.x;
int i_step = blockDim.x;
int k = blockIdx.x;

float *input_k = input + k*inputWindowSize*inputSize;
float *gradOutput_k = gradOutput + k*outputWindowSize*outputSize;
float *inputIndice_k = inputIndice + k*inputWindowSize;
float *outputIndice_k = outputIndice + k*outputWindowSize;

// loop through blocks
for (int m=0; m<outputWindowSize; m++)
{
int outputIdx = (int)outputIndice_k[m] - 1;

float *blockGradOutput = gradOutput_k + m*outputSize;
float *blockGradBias = gradBias + outputIdx*outputSize;

for (int j=tx; j<outputSize; j+=i_step)
gradOutputBuffer[j] = blockGradOutput[j]*scale;

__syncthreads(); // needed for some reason

for (int l=0; l<inputWindowSize; l++)
{
int inputIdx = (int)inputIndice_k[l] - 1;

float *blockInput = input_k + l*inputSize;
float *blockGradWeight = gradWeight + outputIdx*nInputBlock*outputSize*inputSize + inputIdx*outputSize*inputSize;

// addr weights (scalar-products)
for (int i=tx; i<inputSize; i+=i_step)
{
// copy input to buffer
buffer[tx] = blockInput[i];

// multiply accumulate weights
for (int j=0; j<outputSize; j++)
atomicAdd(&(blockGradWeight[j*inputSize + i]), gradOutputBuffer[j]*buffer[tx]);
}
}

__syncthreads(); // needed for some reason

// multiply accumulate biases
for (int j=tx; j<outputSize; j+=i_step)
atomicAdd(&(blockGradBias[j]), gradOutputBuffer[j]);
}
}