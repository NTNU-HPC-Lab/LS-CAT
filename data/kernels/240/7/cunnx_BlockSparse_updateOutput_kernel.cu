#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_BlockSparse_updateOutput_kernel( float *output, const float *input, const float *outputIndice, const float *outputScale, const float *bias, int outputSize, int nOutputBlock, int inputWindowSize, int outputWindowSize)
{
__shared__ float buffer[BLOCKSPARSE_THREADS];
int tx = threadIdx.x;
int i_step = blockDim.x;
int k = blockIdx.x;

float *output_k = output + k*outputWindowSize*outputSize;
const float *input_k = input + k*inputWindowSize*outputWindowSize*outputSize;
const float *outputIndice_k = outputIndice + k*outputWindowSize;
const float *outputScale_k = outputScale + k*outputWindowSize;

for (int m=0; m<outputWindowSize; m++)
{
int outputIdx = (int)outputIndice_k[m] - 1;
float outputScale = outputScale_k[m];

for (int j=tx; j<outputSize; j+=i_step)
{
buffer[tx] = bias[outputIdx*outputSize + j];

for (int l=0; l<inputWindowSize; l++)
buffer[tx] += input_k[l*outputWindowSize*outputSize + m*outputSize + j];

output_k[m*outputSize + j] = outputScale*buffer[tx];
}
}
}