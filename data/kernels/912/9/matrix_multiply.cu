#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply(float *a, float *b, float *c, int num, size_t width)
{
// create shorthand names for threadIdx & blockIdx
int tx = threadIdx.x, ty = threadIdx.y;
int bx = blockIdx.x, by = blockIdx.y;

// allocate 2D tiles in __shared__ memory
__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

// calculate the row & column index of the element
int row = by * blockDim.y + ty;
int col = bx * blockDim.x + tx;

float result = 0;

// loop over the tiles of the input in phases
for(int i = 0; i < (width - 1)/TILE_WIDTH + 1; ++i)
{
// collaboratively load tiles into __shared__
if (row < width && i*TILE_WIDTH + tx < width)
{
s_a[ty][tx] = a[row*width + i*TILE_WIDTH + tx];
}
else
{
s_a[ty][tx] = 0.0;
}
if (col < width && i*TILE_WIDTH + ty < width)
{
s_b[ty][tx] = b[(i*TILE_WIDTH + ty)*width + col];
}
else
s_b[ty][tx] = 0.0;


// wait until all data is loaded before allowing any thread in this block to continue
__syncthreads();

// do dot product between row of s_a and column of s_b
for(int k = 0; k < TILE_WIDTH; ++k)
{
result += s_a[ty][k] * s_b[k][tx];
}

// wait until all threads are finished with the data before allowing any thread in this block to continue
__syncthreads();
}

if (row < width && col < width)
{
c[row*num + col] = result;
}
}