#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int n)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < n)
{
C[i] = A[i] + B[i];
}
}