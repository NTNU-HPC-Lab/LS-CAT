#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecSum(double* devIn, int pow_step, int n)
{
//The thread ID (including its block ID)
int i = blockIdx.x * blockDim.x + threadIdx.x;

//Safety check to prevent unwanted threads.
if(pow_step*i < n)
//The two 'adjacent' elements of the array (or
//the two children in the segment tree) are added and
//the result is stored in the first element.
devIn[pow_step*i] = devIn[pow_step*i+(pow_step/2)] + devIn[pow_step*i];
}