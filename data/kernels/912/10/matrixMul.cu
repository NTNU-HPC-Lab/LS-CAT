#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *A, int *B, int *C, int n)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int num = n;

if (row < num && col < num)
{
long Cvalue = 0;
for (int i = 0; i < num; i++)
{
Cvalue += A[row * num + i] * B[i * num + col];
}
C[row * num + col] = Cvalue;
}
}