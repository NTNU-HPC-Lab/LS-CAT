#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= n) return;
int b;
float sum = 0;
for(b = 0; b < batch; ++b){
int i = b*n + index;
sum += delta[i];
}
bias_updates[index] += sum;
}