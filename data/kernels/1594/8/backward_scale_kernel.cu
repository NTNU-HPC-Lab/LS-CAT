#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
__shared__ float part[BLOCK];
int i,b;
int filter = blockIdx.x;
int p = threadIdx.x;
float sum = 0;
for(b = 0; b < batch; ++b){
for(i = 0; i < size; i += BLOCK){
int index = p + i + size*(filter + n*b);
sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
}
}
part[p] = sum;
__syncthreads();
if (p == 0) {
for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
}
}