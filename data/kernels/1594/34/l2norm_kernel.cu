#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= N) return;
int b = index / spatial;
int i = index % spatial;
int f;
float sum = 0;
for(f = 0; f < filters; ++f){
int index = b*filters*spatial + f*spatial + i;
sum += powf(x[index], 2);
}
sum = sqrtf(sum);
if(sum == 0) sum = 1;
//printf("%f\n", sum);
for(f = 0; f < filters; ++f){
int index = b*filters*spatial + f*spatial + i;
x[index] /= sum;
dx[index] = (1 - x[index]) / sum;
}
}