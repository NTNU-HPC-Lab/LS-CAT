#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void blob_rearrange_kernel2_1d(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
int xy = blockIdx.x*blockDim.x + threadIdx.x;
if(xy>=widthheight)
return;

int ch = blockIdx.y;
int n  = blockIdx.z;


float value=in[(n*channels+ch)*widthheight+xy];

__syncthreads();

int xpad  = (xy % width + padding);
int ypad  = (xy / width + 0);
int xypad = ypad * (width+2*padding) + xpad;

out[(n*pwidthheight+xypad)*channels + ch] = value;
}