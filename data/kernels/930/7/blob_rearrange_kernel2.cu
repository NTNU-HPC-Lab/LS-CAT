#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blob_rearrange_kernel2(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
int xy = blockIdx.x*blockDim.x + threadIdx.x;
if(xy>=widthheight)
return;

int ch = blockIdx.y;
int n  = blockIdx.z;


float value=in[(n*channels+ch)*widthheight+xy];

__syncthreads();

int xpad  = (xy % width + padding);
int ypad  = (xy / width + padding);
int xypad = ypad * (width+2*padding) + xpad;

out[(n*pwidthheight+xypad)*channels + ch] = value;
}