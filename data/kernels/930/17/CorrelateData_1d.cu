#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CorrelateData_1d(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
extern __shared__ char patch_data_char[];

float *patch_data = (float *)patch_data_char;

// First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
int x1 = blockIdx.x*stride1 + max_displacement;
int y1 = blockIdx.y*stride1;
int item = blockIdx.z;
int ch_off = threadIdx.x;

// Load 3D patch into shared shared memory
for(int j = 0; j < kernel_size; j++) { // HEIGHT
for(int i = 0; i < kernel_size; i++) { // WIDTH
int ji_off = ((j * kernel_size) + i) * bottomchannels;
for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
int idxPatchData = ji_off + ch;
patch_data[idxPatchData] = bottom0[idx1];
}
}
}

__syncthreads();

__shared__ float sum[WARPS_PER_BLOCK*THREADS_PER_WARP];

// Compute correlation
for(int top_channel = 0; top_channel < topchannels; top_channel++) {
sum[ch_off] = 0;

int s2o = (top_channel % neighborhood_grid_width + x_shift) * stride2;

for(int j = 0; j < kernel_size; j++) { // HEIGHT
for(int i = 0; i < kernel_size; i++) { // WIDTH
int ji_off = ((j * kernel_size) + i) * bottomchannels;
for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
int x2 = x1 + s2o;

int idxPatchData = ji_off + ch;
int idx2 = ((item * bottomheight + y1+j) * bottomwidth + x2+i) * bottomchannels + ch;
//int idx2 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;

//printf("x1 %d x2 %d bh %d bw %d bc %d i %d ch %d y1 %d idx2 %d\n", x1, x2, bottomheight, bottomwidth, bottomchannels, item, ch, y1, idx2);

sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
}
}
}

__syncthreads();

if(ch_off == 0) {
float total_sum = 0;
for(int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
total_sum += sum[idx];
}
//printf("ch_off %d sum %f\n", ch_off, total_sum);
const int sumelems = kernel_size*kernel_size*bottomchannels;
const int index = ((top_channel*topheight + blockIdx.y)*topwidth)+blockIdx.x;
top[index + item*topcount] = total_sum / (float)sumelems;
}
}


// Aggregate
}