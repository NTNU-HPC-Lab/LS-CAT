#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CorrelateData(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
extern __shared__ char patch_data_char[];

float *patch_data = (float *)patch_data_char;

// First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
int x1 = blockIdx.x*stride1 + max_displacement;
int y1 = blockIdx.y*stride1 + max_displacement;
int item = blockIdx.z;
int ch_off = threadIdx.x;

// Load 3D patch into shared shared memory
for(int j = 0; j < kernel_size; j++) { // HEIGHT
for(int i = 0; i < kernel_size; i++) { // WIDTH
int ji_off = ((j * kernel_size) + i) * bottomchannels;
for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
int idxPatchData = ji_off + ch;
patch_data[idxPatchData] = bottom0[idx1];
}
}
}

__syncthreads();

__shared__ float sum[WARPS_PER_BLOCK*THREADS_PER_WARP];

// Compute correlation
for(int top_channel = 0; top_channel < topchannels; top_channel++) {
sum[ch_off] = 0;

int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;

for(int j = 0; j < kernel_size; j++) { // HEIGHT
for(int i = 0; i < kernel_size; i++) { // WIDTH
int ji_off = ((j * kernel_size) + i) * bottomchannels;
for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
int x2 = x1 + s2o;
int y2 = y1 + s2p;

int idxPatchData = ji_off + ch;
int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + ch;

sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
}
}
}

__syncthreads();

if(ch_off == 0) {
float total_sum = 0;
for(int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
total_sum += sum[idx];
}
const int sumelems = kernel_size*kernel_size*bottomchannels;
const int index = ((top_channel*topheight + blockIdx.y)*topwidth)+blockIdx.x;
top[index + item*topcount] = total_sum / (float)sumelems;
}
}


// Aggregate
}