#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2DBlock2DGrid(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = iy * nx + ix;

if (ix < nx && iy < ny)
MatC[idx] = MatA[idx] + MatB[idx];
}