#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N)   C[i] = A[i] + B[i];
}