#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex() {
printf("blockIdx:(%d, %d, %d) threadIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
blockIdx.x, blockIdx.y, blockIdx.z,
threadIdx.x, threadIdx.y, threadIdx.z,
blockDim.x, blockDim.y, blockDim.z,
gridDim.x, gridDim.y, gridDim.z);
}