#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright: (C) 2017 VisLab, Institute for Systems and Robotics,
*                Instituto Superior Técnico, Universidade de Lisboa, Lisbon, Portugal
* Author: Pedro Vicente <pvicente@isr.tecnico.ulisboa.pt>
* CopyPolicy: Released under the terms of the GNU GPL v3.0.
*
*/



texture<float4, 2, hipReadModeElementType> inTex;

// round up n/m
__global__ void CuDeviceArrayCopyFromTexture( float3* dst, int dstStep, int width, int height )
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if ( x > width || y > height ) return;

float4 res = tex2D(inTex, x, y);
float3* row_y = (float3*)((char*)dst + y * dstStep);
row_y[x] = make_float3(res.x, res.y, res.z);
}