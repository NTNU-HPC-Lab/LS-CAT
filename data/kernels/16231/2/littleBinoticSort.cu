#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void swap(int &a, int &b){
int t = a;
a = b;
b = t;
}
__global__ void littleBinoticSort(int* arr,int num, int numMax){
unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

if(tid >= num) arr[tid] = INT_MAX;

__syncthreads();

for(unsigned int i=2; i<=numMax; i<<=1){
for(unsigned int j=i>>1; j>0; j>>=1){
unsigned int swapIdx = tid ^ j;

if(swapIdx > tid){
if((tid & i)==0){
if(arr[tid] > arr[swapIdx]){
swap(arr[tid], arr[swapIdx]);
}
}
else{
if(arr[tid] < arr[swapIdx]){
swap(arr[tid], arr[swapIdx]);
}
}
}

__syncthreads();
}
}
}