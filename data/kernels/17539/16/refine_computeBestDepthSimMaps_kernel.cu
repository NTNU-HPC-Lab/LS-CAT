#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
//    float floatDepth = depths.y;
float outDepth = -1.0f;

// subpixel refinement
// subpixel refine by Stereo Matching with Color-Weighted Correlation, Hierarchical Belief Propagation, and
// Occlusion Handling Qingxiong pami08
// quadratic polynomial interpolation is used to approximate the cost function between three discrete depth
// candidates: d, dA, and dB.
// TODO: get formula back from paper as it has been lost by encoding.
// d is the discrete depth with the minimal cost, dA ? d A 1, and dB ? d B 1. The cost function is approximated as f?x? ? ax2
// B bx B c.

float simM1 = sims.x;
float simP1 = sims.z;
float sim1 = sims.y;
simM1 = (simM1 + 1.0f) / 2.0f;
simP1 = (simP1 + 1.0f) / 2.0f;
sim1 = (sim1 + 1.0f) / 2.0f;

if((simM1 > sim1) && (simP1 > sim1))
{
float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

float floatDepthM1 = depths.x;
float floatDepthP1 = depths.z;

//-1 : floatDepthM1
// 0 : floatDepth
//+1 : floatDepthP1
// linear function fit
// f(x)=a*x+b
// floatDepthM1=-a+b
// floatDepthP1= a+b
// a = b - floatDepthM1
// floatDepthP1=2*b-floatDepthM1
float b = (floatDepthP1 + floatDepthM1) / 2.0f;
float a = b - floatDepthM1;

outDepth = a * dispStep + b;
};

return outDepth;
}
__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if((x < width) && (y < height))
{
float3 depths = idpts[y * idpts_p + x];
float3 sims = isims[y * isims_p + x];

float outDepth = ((sims.x < sims.y) ? depths.x : depths.y);
float outSim = ((sims.x < sims.y) ? sims.x : sims.y);
outDepth = ((sims.z < outSim) ? depths.z : outDepth);
outSim = ((sims.z < outSim) ? sims.z : outSim);

float refinedDepth = refineDepthSubPixel(depths, sims);
if(refinedDepth > 0.0f)
{
outDepth = refinedDepth;
};

osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
};
}