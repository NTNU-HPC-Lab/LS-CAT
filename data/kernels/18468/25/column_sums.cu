#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void column_sums(const float *A, float *sums, size_t ds){
int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
if (idx < ds){
float sum = 0.0f;
for (size_t i = 0; i < ds; i++)
sum += A[idx+ds*i];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
sums[idx] = sum;
}}