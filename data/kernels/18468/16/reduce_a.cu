#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_a(float *gdata, float *out){
__shared__ float sdata[BLOCK_SIZE];
int tid = threadIdx.x;
sdata[tid] = 0.0f;
size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

while (idx < N) {  // grid stride loop to load data
sdata[tid] += gdata[idx];
idx += gridDim.x*blockDim.x;
}

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
__syncthreads();
if (tid < s)  // parallel sweep reduction
sdata[tid] += sdata[tid + s];
}
if (tid == 0) atomicAdd(out, sdata[0]);
}