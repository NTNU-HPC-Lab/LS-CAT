#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vadd(const float *A, const float *B, float *C, int ds){

int idx = threadIdx.x+blockDim.x*blockIdx.x;
if (idx < ds)
C[idx] = A[idx] + B[idx];
}