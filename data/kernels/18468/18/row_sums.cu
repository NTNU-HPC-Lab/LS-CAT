#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void row_sums(const float *A, float *sums, size_t ds){

int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
if (idx < ds){
float sum = 0.0f;
for (size_t i = 0; i < ds; i++)
sum += A[idx*ds+i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
sums[idx] = sum;
}}