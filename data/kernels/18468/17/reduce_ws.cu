#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_ws(float *gdata, float *out){
__shared__ float sdata[32];
int tid = threadIdx.x;
int idx = threadIdx.x+blockDim.x*blockIdx.x;
float val = 0.0f;
unsigned mask = 0xFFFFFFFFU;
int lane = threadIdx.x % warpSize;
int warpID = threadIdx.x / warpSize;
while (idx < N) {  // grid stride loop to load
val += gdata[idx];
idx += gridDim.x*blockDim.x;
}

// 1st warp-shuffle reduction
for (int offset = warpSize/2; offset > 0; offset >>= 1)
val += __shfl_down_sync(mask, val, offset);
if (lane == 0) sdata[warpID] = val;
__syncthreads(); // put warp results in shared mem

// hereafter, just warp 0
if (warpID == 0){
// reload val from shared mem if warp existed
val = (tid < blockDim.x/warpSize)?sdata[lane]:0;

// final warp-shuffle reduction
for (int offset = warpSize/2; offset > 0; offset >>= 1)
val += __shfl_down_sync(mask, val, offset);

if  (tid == 0) atomicAdd(out, val);
}
}