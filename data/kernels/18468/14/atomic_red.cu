#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomic_red(const float *gdata, float *out){
size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
if (idx < N) atomicAdd(out, gdata[idx]);
}