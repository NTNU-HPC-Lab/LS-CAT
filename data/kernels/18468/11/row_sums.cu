#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void row_sums(const float *A, float *sums, size_t ds){

int idx = blockIdx.x; // our block index becomes our row indicator
if (idx < ds){
__shared__ float sdata[block_size];
int tid = threadIdx.x;
sdata[tid] = 0.0f;
size_t tidx = tid;

while (tidx < ds) {  // block stride loop to load data
sdata[tid] += A[idx*ds+tidx];
tidx += blockDim.x;
}

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
__syncthreads();
if (tid < s)  // parallel sweep reduction
sdata[tid] += sdata[tid + s];
}
if (tid == 0) sums[idx] = sdata[0];
}
}