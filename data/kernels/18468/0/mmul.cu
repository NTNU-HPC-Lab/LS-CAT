#include "hip/hip_runtime.h"
#include "includes.h"

// these are just for timing measurments

// error checking macro
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

if ((idx < ds) && (idy < ds)){
float temp = 0;
for (int i = 0; i < ds; i++)
temp += A[idy*ds+i] * B[i*ds+idx];   // dot product of row and column
C[idy*ds+idx] = temp;
}
}