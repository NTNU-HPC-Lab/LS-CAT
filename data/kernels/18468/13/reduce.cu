#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(float *gdata, float *out, size_t n){
__shared__ float sdata[BLOCK_SIZE];
int tid = threadIdx.x;
sdata[tid] = 0.0f;
size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

while (idx < n) {  // grid stride loop to load data
sdata[tid] = max(gdata[idx], sdata[tid]);
idx += gridDim.x*blockDim.x;
}

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
__syncthreads();
if (tid < s)  // parallel sweep reduction
sdata[tid] = max(sdata[tid + s], sdata[tid]);
}
if (tid == 0) out[blockIdx.x] = sdata[0];
}