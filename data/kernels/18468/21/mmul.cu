#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

// declare cache in shared memory
__shared__ float As[block_size][block_size];
__shared__ float Bs[block_size][block_size];

int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

if ((idx < ds) && (idy < ds)){
float temp = 0;
for (int i = 0; i < ds/block_size; i++) {

// Load data into shared memory
As[threadIdx.y][threadIdx.x] = A[idy * ds + (i * block_size + threadIdx.x)];
Bs[threadIdx.y][threadIdx.x] = B[(i * block_size + threadIdx.y) * ds + idx];

// Synchronize
__syncthreads();

// Keep track of the running sum
for (int k = 0; k < block_size; k++)
temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // dot product of row and column
__syncthreads();

}

// Write to global memory
C[idy*ds+idx] = temp;
}
}