#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vadd(const float *A, const float *B, float *C, int ds){

for (int idx = threadIdx.x+blockDim.x*blockIdx.x; idx < ds; idx+=gridDim.x*blockDim.x)         // a grid-stride loop
C[idx] = A[idx] + B[idx]; // do the vector (element) add here
}