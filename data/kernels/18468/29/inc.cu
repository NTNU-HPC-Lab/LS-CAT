#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inc(int *array, size_t n){
size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
while (idx < n){
array[idx]++;
idx += blockDim.x*gridDim.x; // grid-stride loop
}
}