#include "hip/hip_runtime.h"
#include "includes.h"

/* kernel.cu */




__global__ void AddVector( int vecSize, const float* vecA, const float* vecB, float* vecC)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < vecSize)
vecC[i] = vecA[i] + vecB[i];
}