#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_xy)
{
int x, y, t, z, w;


w = ii % d4;
ii = ii/d4;
z = ii % d3;
ii = ii/d3;
t = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
w = w/scale_factor_xy;
z = z/scale_factor_xy;
t = t/scale_factor_t;
d2 /= scale_factor_t;
d3 /= scale_factor_xy;
d4 /= scale_factor_xy;
return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}
__global__ void upscale(float *input, float *output, long no_elements, int scale_factor_t, int scale_factor_xy, int d1, int d2, int d3, int d4)
{
// output offset:
long ii = threadIdx.x + blockDim.x * blockIdx.x;
ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
if (ii >= no_elements) return;
int ipidx = translate_idx(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_xy);
output[ii]=input[ipidx];
}