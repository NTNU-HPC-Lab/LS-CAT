#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_xy, int off_time, int off_x, int off_y)
{
/* d1 = channel
d2 = time
d3, d4 = height, width
*/
int x, y, t, z, w;
w = ii % d4;
ii = ii/d4;
z = ii % d3;
ii = ii/d3;
t = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
t = t*scale_factor_t+off_time;
w = w*scale_factor_xy+off_x;
z = z*scale_factor_xy+off_y;
d2 *= scale_factor_t;
d3 *= scale_factor_xy;
d4 *= scale_factor_xy;
return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}
__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_xy)
{
int x, y, t, z, w;


w = ii % d4;
ii = ii/d4;
z = ii % d3;
ii = ii/d3;
t = ii % d2;
ii = ii/d2;
y = ii % d1;
ii = ii/d1;
x = ii;
w = w/scale_factor_xy;
z = z/scale_factor_xy;
t = t/scale_factor_t;
d2 /= scale_factor_t;
d3 /= scale_factor_xy;
d4 /= scale_factor_xy;
return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}
__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor_t, int scale_factor_xy, int d1, int d2, int d3, int d4)
{
// output offset:
long ii = threadIdx.x + blockDim.x * blockIdx.x;
ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
if (ii >= no_elements) return;
for (int i=0; i < scale_factor_t; i++){
for(int j=0; j < scale_factor_xy; j++){
for(int k=0; k < scale_factor_xy; k++){
int ipidx = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_xy, i, j, k);
gradInput_data[ii] += gradOutput_data[ipidx];
}
}
}
}