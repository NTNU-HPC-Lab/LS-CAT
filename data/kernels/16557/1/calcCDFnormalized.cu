#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcCDFnormalized(const unsigned int *histo, float *cdf, size_t width, size_t height) {
for (int i = 0; i <= threadIdx.x; i++) {
cdf[threadIdx.x] += (float) histo[i];
}
cdf[threadIdx.x] *= 1.0f / float((width * height));
}