#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcCDF(float *cdf, unsigned int *histo, int imageWidth, int imageHeight, int length) {

__shared__ float partialScan[SIZE_CDF];
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < SIZE_CDF && i < 256) {
partialScan[i] = (float) histo[i] / (float) (imageWidth * imageHeight);

}
__syncthreads();

for (unsigned int stride = 1; stride <= SIZE_HISTO; stride *= 2) {
unsigned int index = (threadIdx.x + 1) * stride * 2 - 1;
if (index < SIZE_CDF && index < length)
partialScan[index] += partialScan[index - stride];
__syncthreads();
}

for (unsigned int stride = SIZE_HISTO / 2; stride > 0; stride /= 2) {
__syncthreads();
unsigned int index = (threadIdx.x + 1) * stride * 2 - 1;
if (index + stride < SIZE_CDF && index + stride < length) {
partialScan[index + stride] += partialScan[index];
}
}

__syncthreads();
if (i < SIZE_CDF && i < 256) {
cdf[i] += partialScan[threadIdx.x];
}
}