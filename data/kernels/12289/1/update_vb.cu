#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_vb(float *d_verts_ptr, int vertex_count, float timeElapsed)
{
const unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;

if (threadId < vertex_count * 4)
{
float valx = d_verts_ptr[threadId * 4 + 0];
float valy = d_verts_ptr[threadId * 4 + 1];
float valz = d_verts_ptr[threadId * 4 + 2];


d_verts_ptr[threadId * 4 + 0] = valx * timeElapsed;
d_verts_ptr[threadId * 4 + 1] = valy * timeElapsed;
d_verts_ptr[threadId * 4 + 2] = valz * timeElapsed;
}
}