#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {





}
__global__ void vmul(const float *a, const float *b, float *c)
{
int i = blockIdx.x *blockDim.x + threadIdx.x;
c[i] = a[i] * b[i];
}