#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
const int threads = BLOCK;
__shared__ float local[threads];

int id = threadIdx.x;
local[id] = 0;

int filter = blockIdx.x;

int i, j;
for(j = 0; j < batch; ++j){
for(i = 0; i < spatial; i += threads){
int index = j*spatial*filters + filter*spatial + i + id;
local[id] += (i+id < spatial) ? x[index] : 0;
}
}
__syncthreads();

if(id == 0){
mean[filter] = 0;
for(i = 0; i < threads; ++i){
mean[filter] += local[i];
}
mean[filter] /= spatial * batch;
}
}