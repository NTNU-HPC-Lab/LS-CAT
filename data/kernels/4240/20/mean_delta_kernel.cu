#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= filters) return;
int j,k;
mean_delta[i] = 0;
for (j = 0; j < batch; ++j) {
for (k = 0; k < spatial; ++k) {
int index = j*filters*spatial + i*spatial + k;
mean_delta[i] += delta[index];
}
}
mean_delta[i] *= (-1.F/sqrtf(variance[i] + .000001f));
}