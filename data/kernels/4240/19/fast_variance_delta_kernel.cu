#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
const int threads = BLOCK;
__shared__ float local[threads];

int id = threadIdx.x;
local[id] = 0;

int filter = blockIdx.x;

int i, j;
for(j = 0; j < batch; ++j){
for(i = 0; i < spatial; i += threads){
int index = j*spatial*filters + filter*spatial + i + id;

local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
}
}
__syncthreads();

if(id == 0){
variance_delta[filter] = 0;
for(i = 0; i < threads; ++i){
variance_delta[filter] += local[i];
}
variance_delta[filter] *= -.5 * powf(variance[filter] + .000001f, (float)(-3./2.));
}
}