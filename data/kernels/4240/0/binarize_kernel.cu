#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}


__global__ void binarize_kernel(float *x, int n, float *binary)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= n) return;
binary[i] = (x[i] >= 0) ? 1 : -1;
}