#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************************************
*
*******************************************************************************/

/*************************************************************************

/*************************************************************************/

/*************************************************************************/
__global__ void drawColor(unsigned char* optr, const float* red, const float* green, const float* blue) {
// map from threadIdx/BlockIdx to pixel position
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

float theRed = red[offset];
//  theRed = (theRed / 50.0) + 0.5;
if (theRed < 0) theRed = 0;
if (theRed > 1) theRed = 1;

float theGreen = green[offset];
//  theGreen = (theGreen / 50.0) + 0.5;
if (theGreen < 0) theGreen = 0;
if (theGreen > 1) theGreen = 1;

float theBlue = blue[offset];
//  theBlue = (theBlue / 50.0) + 0.5;
if (theBlue < 0) theBlue = 0;
if (theBlue > 1) theBlue = 1;


optr[offset * 4 + 0] = 255 * theRed;    // red
optr[offset * 4 + 1] = 255 * theGreen;  // green
optr[offset * 4 + 2] = 255 * theBlue;   // blue
optr[offset * 4 + 3] = 255;             // alpha (opacity)
}