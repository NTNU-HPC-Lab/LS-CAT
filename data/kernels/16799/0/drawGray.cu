#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************************************
*
*******************************************************************************/

/*************************************************************************

/*************************************************************************/

/*************************************************************************/
__global__ void drawGray(unsigned char* optr, const float* outSrc) {
// map from threadIdx/BlockIdx to pixel position
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

float val = outSrc[offset];

val = (val / 50.0) + 0.5; //get {-25 to 25} range into {0 to 1} range
if (val < 0) val = 0;
if (val > 1) val = 1;

optr[offset * 4 + 0] = 255 * val;       // red
optr[offset * 4 + 1] = 255 * val;       // green
optr[offset * 4 + 2] = 255 * val;       // blue
optr[offset * 4 + 3] = 255;             // alpha (opacity)
}