#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* red, float* green, float* blue, unsigned long N){

int x = threadIdx.x + (blockIdx.x * blockDim.x);
int y = threadIdx.y + (blockIdx.y * blockDim.y);
unsigned long tid = x + (y * blockDim.x * gridDim.x);

if(tid < N){
red[tid] = .5;
blue[tid] = .7;
green[tid]= .2;
}
}