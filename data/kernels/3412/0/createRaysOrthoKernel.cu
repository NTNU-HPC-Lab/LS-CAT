#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createRaysOrthoKernel(float4* rays, int width, int height, float x0, float y0, float z, float dx, float dy, unsigned rayMask )
{
int rayx = threadIdx.x + blockIdx.x*blockDim.x;
int rayy = threadIdx.y + blockIdx.y*blockDim.y;
if( rayx >= width || rayy >= height )
return;

float tMinOrMask = 0.0f;
if( rayMask )
tMinOrMask = __int_as_float( rayMask );

int idx = rayx + rayy*width;
rays[2*idx+0] = make_float4( x0+rayx*dx, y0+rayy*dy, z, tMinOrMask );  // origin, tmin
rays[2*idx+1] = make_float4( 0, 0, 1, 1e34f ); // dir, tmax
}