#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void MatrixMulKernelV3(float* d_M, float* d_N, float* d_P, int Width)
{
__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; // [TILE_WIDTH][TILE_WIDTH]
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; // [TILE_WIDTH][TILE_WIDTH]

int bx = blockIdx.x; int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;
float Pvalue = 0;

for (int ph = 0; ph < ceil(Width/(float)TILE_WIDTH); ++ph){
if ((Row< Width) && (ph*TILE_WIDTH+tx)< Width)
Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];

if ((ph*TILE_WIDTH+ty)<Width && Col<Width)
Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];

__syncthreads();

for (int k = 0; k < TILE_WIDTH; ++k)
Pvalue += Mds[ty][k] * Nds[k][tx];

__syncthreads();
}

if ((Row<Width) && (Col<Width))
d_P[Row*Width + Col] = Pvalue;

}