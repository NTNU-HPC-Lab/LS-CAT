#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imageBlurKernel(unsigned char* d_image, int h, int w)
{
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

int blurSize = 8;

Row = Row * blurSize;
Col = Col * blurSize;

int r, g, b;
int p_r = 0;
int p_g = 0;
int p_b = 0;
int i, j;

if((Row+blurSize < h) && (Col+blurSize < w)){
for(i = 0; i < blurSize; i++)
for(j = 0; j < blurSize; j++){
r = d_image[4 * w * (Row+j) + 4 * (Col+i) + 0];
g = d_image[4 * w * (Row+j) + 4 * (Col+i) + 1];
b = d_image[4 * w * (Row+j) + 4 * (Col+i) + 2];

p_r += r;
p_g += g;
p_b += b;
}

p_r = p_r / (blurSize * blurSize);
p_g = p_g / (blurSize * blurSize);
p_b = p_b / (blurSize * blurSize);

for(i = 0; i < blurSize; i++)
for(j = 0; j < blurSize; j++){
d_image[4 * w * (Row+j) + 4 * (Col+i) + 0] = p_r;
d_image[4 * w * (Row+j) + 4 * (Col+i) + 1] = p_g;
d_image[4 * w * (Row+j) + 4 * (Col+i) + 2] = p_b;
}
}
}