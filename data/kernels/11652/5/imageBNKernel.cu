#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imageBNKernel(unsigned char* d_image, int h, int w)
{
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;
int r, g, b;

if((Row < h) && (Col < w)){
r = d_image[4 * w * Row + 4 * Col + 0];
g = d_image[4 * w * Row + 4 * Col + 1];
b = d_image[4 * w * Row + 4 * Col + 2];

d_image[4 * w * Row + 4 * Col + 0] = 0;
d_image[4 * w * Row + 4 * Col + 1] = 0;
d_image[4 * w * Row + 4 * Col + 2] = 0;
d_image[4 * w * Row + 4 * Col + 3] = (int)(r*0.21 + g*0.71 + b*0.07);
}
}