#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void matrixVectMultKernel(float* A, float* B, float* C, int n)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int elementPos = i;
i = i * n;
int limit = i + n;
int j = 0;

if(i < n*n){
C[elementPos] = 1;

while((i < limit) && (j < n)){
C[elementPos] = C[elementPos] * (A[i]+B[j]);
i++;
j++;
}
}
}