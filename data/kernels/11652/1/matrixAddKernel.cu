#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void matrixAddKernel(float* A, float* B, float* C, int n)
{
int Row = blockIdx.y * blockDim.y + threadIdx.y;
int Col = blockIdx.x * blockDim.x + threadIdx.x;

if((Row < n) && (Col < n))
C[Row * n + Col] = A[Row * n + Col] + B[Row * n + Col];
}