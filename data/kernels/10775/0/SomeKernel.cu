#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define delta		     10
#define rows			 50
#define columns			 50


__global__ void SomeKernel(int* res, int* data, int col, int row,int y, int step)
{
unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
//Считаем идентификатор текущего потока
int currDelta = 0;
for (int i=step*threadId; (i<(threadId+1)*step) && (i < col); i++) //Работа со столбцами по потокам
{
for (int j = y; j > 0; j--) //Здесь работа со строками
{
currDelta = data[i + j*row] - data[i + (j-1)*row];
//если текущая разность больше дельты, то запоминаем у-координату
if( ( currDelta >= 0 ? currDelta : currDelta*-1 ) > 10){
res[i] = j-1;
break;
}
}
}
}