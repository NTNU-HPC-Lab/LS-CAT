#include "hip/hip_runtime.h"
#include "includes.h"
/******************************************************************************
* © Mathias Bourgoin, Université Pierre et Marie Curie (2011)
*
* Mathias.Bourgoin@gmail.com
*
* This software is a computer program whose purpose is allow GPU programming
* with the OCaml language.
*
* This software is governed by the CeCILL-B license under French law and
* abiding by the rules of distribution of free software.  You can  use,
* modify and/ or redistribute the software under the terms of the CeCILL-B
* license as circulated by CEA, CNRS and INRIA at the following URL
* "http://www.cecill.info".
*
* As a counterpart to the access to the source code and  rights to copy,
* modify and redistribute granted by the license, users are provided only
* with a limited warranty  and the software's author,  the holder of the
* economic rights,  and the successive licensors  have only  limited
* liability.
*
* In this respect, the user's attention is drawn to the risks associated
* with loading,  using,  modifying and/or developing or reproducing the
* software by the user in light of its specific status of free software,
* that may mean  that it is complicated to manipulate,  and  that  also
* therefore means  that it is reserved for developers  and  experienced
* professionals having in-depth computer knowledge. Users are therefore
* encouraged to load and test the software's suitability as regards their
* requirements in conditions enabling the security of their systems and/or
* data to be ensured and,  more generally, to use and operate it in the
* same conditions as regards security.
*
* The fact that you are presently reading this means that you have had
* knowledge of the CeCILL-B license and that you accept its terms.
*
* NOTE:  This file contains source code provided by NVIDIA Corporation.
*******************************************************************************/
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif
__global__ void int_bubble_filter( int* input, const int* vec1, int* output, const int count)
{
int i;
int k = 1;
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid <= count/2)
{
output[tid*2] = vec1[tid*2];
output[tid*2+1] = vec1[tid*2+1];
//barrier(CLK_GLOBAL_MEM_FENCE);

for (int n = 0; n < count*2; n++)
{
k = (k)?0:1;
i = (tid*2) + k;
if( i+1 < count)
{
if ((!input[i]) && (input[i+1]))
{
input[i] = 1;
input[i+1] = 0;
output[i] = output[i+1];
output[i+1] = 0;
}
else
{
if (!input[i])
output[i] = 0;
if (!input[i+1])
output[i+1] = 0;
}
}
__syncthreads();
}
}
}