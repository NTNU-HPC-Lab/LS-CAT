#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

struct point{
float x;
float y;
};




struct point2{
double x;
double y;
};




#ifdef __cplusplus
}
#endif
__global__ void pi(const struct point* A, int* res, const int nbPoint, const float ray){
const int idx = 32*blockDim.x * blockIdx.x + threadIdx.x;
if (idx < nbPoint-32*blockDim.x)
#pragma unroll 16
for (int j = 0; j < 32; j++) {
int i = idx + blockDim.x * j;
res[i] = (A[i].x*A[i].x + A[i].y*A[i].y <= ray);
}
}