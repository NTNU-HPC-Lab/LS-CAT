#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgbToGreyKernel(int height,int width ,unsigned char *input_img, unsigned char *output_img)
{
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

if(row<height && col<width)
{
int idx = row*width + col;
float red = (float)input_img[3*idx];
float green = (float)input_img[3*idx+1];
float blue = (float)input_img[3*idx+2];

output_img[idx] = 0.21*red + 0.71*green + 0.07*blue;
}

}