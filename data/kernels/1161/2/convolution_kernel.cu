#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_kernel(unsigned char *input_img, unsigned char *output_img, int height, int width)
{

__shared__ unsigned char input_shared[W][W]; //Shared Memory required for a tile and its halo elements(3 channels)

int chan;
for(chan=0;chan<3;chan++)    //3 Channel Image
{
int tx = threadIdx.x;
int ty = threadIdx.y;

int output_row = blockIdx.x*TILE_WIDTH + tx;
int output_col = blockIdx.y*TILE_WIDTH + ty;

int input_row = output_row - MASK_WIDTH/2;
int input_col = output_col - MASK_WIDTH/2;

if((input_row >= 0) && (input_row < height) && (input_col >= 0) && (input_col < width))
{
input_shared[tx][ty] = input_img[(input_row*width + input_col)*3 + chan];
}
else
{
if(input_row<0 && input_col<0)
{
input_shared[tx][ty] = input_img[chan];
}
else if(input_row<0 && input_col<width)
{
input_shared[tx][ty] = input_img[3*input_col + chan];
}
else if(input_row<0)
{
input_shared[tx][ty] = input_img[3*(width-1) + chan];
}
else if(input_row<height && input_col<0)
{
input_shared[tx][ty] = input_img[input_row*width*3 + chan];
}
else if(input_row<height && input_col>width)
{
input_shared[tx][ty] = input_img[(input_row*width +width-1)*3 + chan];
}
else if(input_row>height && input_col<0)
{
input_shared[tx][ty] = input_img[width*(height-1)*3 + chan];
}
else if(input_row>height && input_col<width)
{
input_shared[tx][ty] = input_img[(width*(height-1)+input_col)*3 + chan];
}
else
{
input_shared[tx][ty] = input_img[(width*(height-1) + (width-1))*3 + chan];
}
}

__syncthreads();

int i;
if(tx<TILE_WIDTH && ty<TILE_WIDTH)
{
int j;
int freq[256];

for(i=0;i<256;i++)freq[i]=0;

for(i=0;i<MASK_WIDTH;i++)
{
for(j=0;j<MASK_WIDTH;j++)
{
freq[input_shared[tx+i][ty+j]]++;
}
}
j=0;
for(i=0;i<256;i++)
{
j=j+freq[i];
if(j>((MASK_WIDTH*MASK_WIDTH)/2))break;
}
}

if(output_row<height && output_col<width)
{
output_img[(output_row*width + output_col)*3 + chan] = i;
}
__syncthreads();
}

}