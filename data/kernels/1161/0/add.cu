#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void add(int N, double *a,double *b, double *c)
{
int tid = blockIdx.x*blockDim.x + threadIdx.x;
if(tid < N)
{
c[tid] = a[tid]+b[tid];
}

}