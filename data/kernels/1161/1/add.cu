#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int N, double *a,double *b)
{
int tid = blockIdx.x*blockDim.x + threadIdx.x;
if(tid < N)
{
b[tid] = a[tid]*a[tid];
}

}