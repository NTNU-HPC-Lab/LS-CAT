#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void padding_nm2v( float *nm2v_re, float *nm2v_im, int nfermi, int norbs, int nvirt, int vstart)
{
int i = blockIdx.x * blockDim.x + threadIdx.x; //nocc
int j = blockIdx.y * blockDim.y + threadIdx.y; //nvirt

if (i > vstart && i < nfermi)
{
if ( j < norbs - vstart )
{
nm2v_re[i*nvirt + j] = 0.0;
nm2v_im[i*nvirt + j] = 0.0;
}
}

}