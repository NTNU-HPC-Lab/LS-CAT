#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_energy_gpu(float *ksn2e, float *ksn2f, double omega_re, double omega_im, float *nm2v_re, float *nm2v_im, int nfermi, int norbs, int nvirt, int vstart)
{
int i = blockIdx.x * blockDim.x + threadIdx.x; //nocc
int j = blockIdx.y * blockDim.y + threadIdx.y; //nvirt
float en=0.0, fn=0.0, em=0.0, fm=0.0, old_re, old_im;
double d1p, d1pp, d2p, d2pp, alpha, beta;

if (i < nfermi)
{
en = ksn2e[i];
fn = ksn2f[i];
if ( j < norbs - vstart )
{
em = ksn2e[j + vstart];
fm = ksn2f[j + vstart];

d1p = omega_re - (em-en); d1pp = omega_im;
d2p = omega_re + (em-en); d2pp = omega_im;

alpha = d1p/(d1p*d1p + d1pp*d1pp) - d2p/(d2p*d2p + d2pp*d2pp);
beta = -d1pp/(d1p*d1p + d1pp*d1pp) + d2pp/(d2p*d2p + d2pp*d2pp);
old_re = nm2v_re[i*nvirt + j];
old_im = nm2v_im[i*nvirt + j];

nm2v_re[i*nvirt + j] = (fn - fm)*(old_re*alpha - old_im*beta);
nm2v_im[i*nvirt + j] = (fn - fm)*(old_re*beta + old_im*alpha);
//printf("i = %d, j = %d, m = %d, alpha = %f, beta = %f, old_re = %f, old_im = %f, nm2v_re = %f, nm2v_im = %f\n",
//    i, j, m, alpha, beta, old_re, old_im, nm2v_re[index], nm2v_im[index]);

//nm2v = nm2v * (fn-fm) * ( 1.0 / (comega - (em - en)) - 1.0 /(comega + (em - en)) );
}
}
}