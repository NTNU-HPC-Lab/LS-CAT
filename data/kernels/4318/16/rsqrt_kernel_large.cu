#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rsqrt_kernel_large(float* x, unsigned int len, unsigned int rowsz) {
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
if (idx < len) x[idx] = x[idx] > 0 ? rsqrt(x[idx]) : 0;
}