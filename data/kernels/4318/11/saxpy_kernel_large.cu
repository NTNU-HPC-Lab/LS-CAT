#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_kernel_large(const float a, const float* x, const float* y, float* result, unsigned int len, unsigned int rowsz) {
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
if (idx < len) result[idx] = a * x[idx] + y[idx];
}