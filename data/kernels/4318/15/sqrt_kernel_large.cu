#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sqrt_kernel_large(float* x, unsigned int len, unsigned int rowsz) {
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
if (idx < len) x[idx] = sqrt(x[idx]);
}