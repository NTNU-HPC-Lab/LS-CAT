#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_kernel(const float a, const float* x, const float* y, float* result, unsigned int len) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) result[idx] = a * x[idx] + y[idx];
}