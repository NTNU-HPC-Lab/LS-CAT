#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_max_kernel(const float* x, int len, int blen, float* result) {
__shared__ float value[256];
int bstart = blen * blockIdx.x;
int start = bstart + threadIdx.x;
int end = min(len, bstart + blen);

float v = 0;
for (int i = start; i < end; i += blockDim.x) v = max(v, fabs(x[i]));
value[threadIdx.x] = v;
// reduce to the first two values
__syncthreads();
if (threadIdx.x < 128)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 128]);
__syncthreads();
if (threadIdx.x < 64)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 64]);
__syncthreads();
if (threadIdx.x < 32)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 32]);
if (threadIdx.x < 16)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 16]);
if (threadIdx.x < 8)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 8]);
if (threadIdx.x < 4)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 4]);
if (threadIdx.x < 2)
value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 2]);
// write back
if (threadIdx.x == 0) result[blockIdx.x] = max(value[0], value[1]);
}