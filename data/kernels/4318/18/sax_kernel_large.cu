#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sax_kernel_large(const float a, const float* x, float* result, unsigned int len, unsigned int rowsz) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;
if (idx < len) result[idx] = a * x[idx];
}