#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vxy_kernel_large(const float* x, float* y, float* result, unsigned int len, unsigned int rowsz) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + rowsz * blockIdx.y;
if (idx < len) result[idx] = x[idx] * y[idx];
}