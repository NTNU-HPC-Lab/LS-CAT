#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vxy_kernel(const float* x, float* y, float* result, unsigned int len) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) result[idx] = x[idx] * y[idx];
}