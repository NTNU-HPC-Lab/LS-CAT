#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sxypz_kernel_large(float a, const float* x, const float* y, const float* z, float* result, unsigned int len, unsigned int rowsz) {
unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
if (idx < len) result[idx] = a * x[idx] * y[idx] + z[idx];
}