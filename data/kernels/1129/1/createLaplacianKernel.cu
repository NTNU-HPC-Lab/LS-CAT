#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createLaplacianKernel(float *grid, float *kernel, int nrDimensions, int nrGridElements) {
size_t x = threadIdx.x + blockDim.x * blockIdx.x;

if (x >= nrGridElements)
return;

for(int d = 0; d < nrDimensions; ++d) {
if (d == 0)
kernel[x] = grid[x];
else
kernel[x] += grid[x + d*nrGridElements];
}
}