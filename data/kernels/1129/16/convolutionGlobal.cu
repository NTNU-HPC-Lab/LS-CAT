#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionGlobal(float *imgIn, float *kernel, float *imgOut, int w, int h, int nc, int kernelSize){
size_t x = threadIdx.x + blockDim.x * blockIdx.x;
size_t y = threadIdx.y + blockDim.y * blockIdx.y;
size_t k = kernelSize;

int r=k/2;

//check for boundarys of the block
if(x>=w || y>=h) return;

//iterate over all channels
for(unsigned int c=0;c<nc;c++) {
float sum=0;
//do convolution
for(unsigned int i=0;i<k;i++){
unsigned int x_new;
//clamping x
if(x+r<i) x_new=0;
else if(x+r-i>=w) x_new=w-1;
else x_new=x+r-i;
for(unsigned int j=0;j<k;j++){
//clamping y
unsigned int y_new;
if(y+r<j)
y_new=0;
else if(y+r-j>=h)
y_new=h-1;
else
y_new=y+r-j;
sum+=kernel[i+j*k]*imgIn[x_new+y_new*w+w*h*c];
}
}
//imgOut[id(x,y,w,h,c)]=sum;
imgOut[x + y*w + c*w*h]=sum;
}
}