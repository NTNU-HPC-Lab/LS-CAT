#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixPolyderNewLayout(const float *coefImg, float *coefImgDer, const int w, const int h, const int m, size_t yOffset){
size_t x = threadIdx.x + blockDim.x * blockIdx.x;
size_t y = threadIdx.y + blockDim.y * blockIdx.y;
if(x >= w || y >= h) return;

size_t xOffsetDer = m-1;
size_t yOffsetDer = w*xOffsetDer;

size_t xOffsetCoef = m;
size_t yOffsetCoef = w*xOffsetCoef;

for (int i = 0; i < m - 1; ++i) //if of degree d=2, we have n=3 coeffs ax'2 + bx +c
{
size_t idxDer = x*xOffsetDer + y*yOffsetDer + i;
size_t idxCoef = x*xOffsetCoef + y*yOffsetCoef + i;

coefImgDer[idxDer]=coefImg[idxCoef]*(m-i-1);
}
}