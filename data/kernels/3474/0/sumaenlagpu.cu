#include "hip/hip_runtime.h"
#include "includes.h"


#define N (4096*4096)
#define HILOS_POR_BLOQUE 512


__global__ void sumaenlagpu(int *a, int *b, int *c, int n){
int index = threadIdx.x + blockIdx.x*blockDim.x;
if (index < n){
c[index] = a[index] + b[index];
}
}