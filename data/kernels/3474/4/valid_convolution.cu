#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void convolution(int conv_col, int conv_row, float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
int conv_index = conv_col+ conv_row*max_col;
d_conv[conv_index] = 0;
for(int k_row = 0;  k_row < k_size; k_row ++){
for(int k_col = 0;  k_col < k_size ; k_col ++){
d_conv[conv_index] +=
d_kernel[k_col + (k_row*k_size)] *
d_matrix[(conv_col+k_col) + (conv_row+k_row)*size_x];
//		printf("row %i col %i d_conv[] = %f \n", row, col, d_conv[col+ row*max_col]);
}
}
}
__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

if(max_row > row && max_col > col){
convolution(col, row, d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
}
}