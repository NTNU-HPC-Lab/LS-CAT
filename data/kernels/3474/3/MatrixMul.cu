#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(float *darray_1, float *darray_2 , float *dres_arr, int n){
// cols and rows definition
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;
// Mat mult operation
for(int i = 0; i<n; i++){
dres_arr[row*n+col]+= darray_1[row*n+i]*darray_2[col+i*n];
// printf("row %i * height %i col %i index %i res %f\n", row, n, col, i, dres_arr[row*n+col]);
}
}