#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void piCalc(double *area, double width, int rects) {
double mid, height;
// Get our index
int index = threadIdx.x + (blockIdx.x * blockDim.x);
// Pos in array
int id = index;
// do while we are inside our array
while(index<rects){
//Original pi algo
mid = (index + 0.5) * width;
height = 4.0 / (1.0 + mid * mid);
area[id] += height;
// Move our index
index += (blockDim.x*gridDim.x);
}
}