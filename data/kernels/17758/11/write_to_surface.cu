#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void write_to_surface(const float *data, hipSurfaceObject_t surface, const int width, const int height) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
const int b = 4 * blockIdx.z;

if (x < width && y < height) {
const int wh = width * height;
const int offset = b * wh + y * width + x;

float4 tmp;
tmp.x = data[0 * wh + offset];
tmp.y = data[1 * wh + offset];
tmp.z = data[2 * wh + offset];
tmp.w = data[3 * wh + offset];

surf2DLayeredwrite<float4>(tmp, surface, x * sizeof(float4), y, blockIdx.z);
}
}