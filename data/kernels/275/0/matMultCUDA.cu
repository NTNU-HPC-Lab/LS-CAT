#include "hip/hip_runtime.h"
#include "includes.h"
// Second CUDA program
// Ping-Che Chen




#define BLOCK_SIZE	16




__global__ static void matMultCUDA(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n)
{
__shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
const int tidc = threadIdx.x;
const int tidr = threadIdx.y;
const int bidc = blockIdx.x * BLOCK_SIZE;
const int bidr = blockIdx.y * BLOCK_SIZE;
int i, j;

float results = 0;
float comp = 0;

for(j = 0; j < n; j += BLOCK_SIZE) {
matA[tidr][tidc] = a[(tidr + bidr) * lda + tidc + j];
matB[tidr][tidc] = b[(tidr + j) * ldb + tidc + bidc];

__syncthreads();

for(i = 0; i < BLOCK_SIZE; i++) {
float t;
comp -= matA[tidr][i] * matB[i][tidc];
t = results - comp;
comp = (t - results) + comp;
results = t;
}

__syncthreads();
}

c[(tidr + bidr) * ldc + tidc + bidc] = results;
}