#include "hip/hip_runtime.h"
#include "includes.h"
#define H 5
#define W 5

using namespace std;

__global__ void mult_dist(int *d_A, int *d_B,int *d_C){
int i = blockIdx.y*blockDim.y+threadIdx.y;//todos los valores fila
int j = blockIdx.x*blockDim.x+threadIdx.x;//todos los valores columna
if(i < H && j < W){
int Pvalue = 0;
for(int k=0; k<H; k++){
Pvalue += d_A[i*W+k] * d_B[k*W+j];
}
d_C[i*W+j] = Pvalue;
}
}