#include "hip/hip_runtime.h"
#include "includes.h"
/*------------------------GPU RANKING----------------------------------------START-------*/

/*------------------------shfl_scan_test-----------------------------------------------Start*/
/*------------------------shfl_scan_test-----------------------------------------------End*/
/*------------------------Final Ranking-----------------------------------------------Start*/

/*------------------------Final_ranking-----------------------------------------------End*/

/*-----------------------GPU RANKING------------------------------------------END--------*/

/*-----------------------iDivUp--------------------------------------------------------Start*/

__global__ void final_ranking(float *data , int *rank , float *partial_data , int *partial_rank , int len)
{
__shared__ float value_buf;
__shared__ int rank_buf;

int id = ((blockIdx.x*blockDim.x)+threadIdx.x);
if(id>len) return;

if(threadIdx.x == 0)
{
value_buf = partial_data[blockIdx.x];
rank_buf = partial_rank[blockIdx.x];
}
__syncthreads();
if(data[id] == value_buf)
{
rank[id] = rank_buf;
}
}