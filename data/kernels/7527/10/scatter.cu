#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter(int *d_array , int *d_predicateArray, int *d_scanArray,int *d_compactedArray, int d_numberOfElements)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index < d_numberOfElements)
{
if(d_predicateArray[index]==1)
{
d_compactedArray[d_scanArray[index]-1] = d_array[index];

}
}
}