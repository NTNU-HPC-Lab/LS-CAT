#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void predicateDevice(int *d_array , int *d_predicateArrry , int d_numberOfElements,int bit,int bitset)
{
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < d_numberOfElements)
{
if(bitset == 0)
{
if((d_array[index] & bit) == 0)
{
d_predicateArrry[index] = 1;
}
else
{
d_predicateArrry[index] = 0;
}
}
else
{
if((d_array[index] & bit) != 0)
{
d_predicateArrry[index] = 1;
}
else
{
d_predicateArrry[index] = 0;
}
}
}
}