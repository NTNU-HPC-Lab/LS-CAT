#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hillisSteeleScanDevice(int *d_predicateArray , int d_numberOfElements ,int *d_tmpArray,int d_offset)
{
int index = blockIdx.x * blockDim.x +  threadIdx.x;
if(index < d_numberOfElements)
{
d_tmpArray[index] = d_predicateArray[index];
if(index - d_offset >= 0)
{

d_tmpArray[index] = d_predicateArray[index] + d_predicateArray[index-d_offset];
}
}
}