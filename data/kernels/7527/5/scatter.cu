#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter(int *d_array , int *d_scanArray , int *d_predicateArrry,int * d_scatteredArray ,int d_numberOfElements,int offset)
{
int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < d_numberOfElements)
{
if(d_predicateArrry[index] == 1)
{
d_scatteredArray[d_scanArray[index] - 1 +offset ] = d_array[index];

}
}
}