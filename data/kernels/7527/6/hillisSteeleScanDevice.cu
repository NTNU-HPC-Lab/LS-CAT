#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray,int moveIndex)
{
int index = threadIdx.x + blockDim.x * blockIdx.x;
if(index > numberOfElements)
{
return;
}
d_tmpArray[index] = d_array[index];
if(index - moveIndex >=0)
{

d_tmpArray[index] = d_tmpArray[index] +d_array[index - moveIndex];
}
}