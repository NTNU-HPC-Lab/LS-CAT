#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
int index = blockIdx.x * blockDim.x + threadIdx.x ;
int sum = 0;

int j=0;
for(int i=index;i<numberOfElements;i = i+(numberOfBlocks*numberOfThreadsPerBlock))
{
sum = sum + d_array[i];
j++;
}
extern __shared__ int d_blockMemmory[];
d_blockMemmory[threadIdx.x] = sum;
sum =0;
__syncthreads();


if(threadIdx.x == 0)
{
for(int i =0; i<numberOfThreadsPerBlock;i++)
{
sum = sum+ d_blockMemmory[i];
}
d_global[blockIdx.x] = sum;
}
}