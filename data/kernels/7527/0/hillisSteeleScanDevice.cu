#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray,int moveIndex)
{
int index = threadIdx.x + blockDim.x * blockIdx.x;
if(index > numberOfElements)
{
return;
}
if(index - moveIndex >=0)
{
d_tmpArray[index] = d_array[index];
d_tmpArray[index] = d_tmpArray[index] +d_array[index - moveIndex];
d_array[index] = d_tmpArray[index];
}


}