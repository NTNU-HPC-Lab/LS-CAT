#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
int index = blockIdx.x * blockDim.x + threadIdx.x ;
index = index * elementsPerThread;

if(index>numberOfElements)
{
return;
}

int sum = 0;

for(int i=index;i<index+elementsPerThread;i++)
{
sum = sum + d_array[i];
}

extern __shared__ int d_blockMemmory[];
d_blockMemmory[threadIdx.x] = sum;
sum =0;
__syncthreads();


if(threadIdx.x == 0)
{
for(int i =0; i<numberOfThreadsPerBlock;i++)
{
sum = sum+ d_blockMemmory[i];
}
d_global[blockIdx.x] = sum;
}
}