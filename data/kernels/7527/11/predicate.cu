#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void predicate(int *d_array, int d_numberOfElements,int *d_predicateArray)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index <d_numberOfElements)
{
if(d_array[index]%32== 0)
{
d_predicateArray[index] =1;
}
else
{
d_predicateArray[index]  = 0;
}
}
}