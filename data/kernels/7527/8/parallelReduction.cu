#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
int index = blockIdx.x * blockDim.x + threadIdx.x ;
int sum = 0;

int j=0;
for(int i=index;i<numberOfElements;i = i+(numberOfBlocks*numberOfThreadsPerBlock))
{
sum = sum + d_array[i];
j++;
}
d_global[index] = sum;
}