#include "hip/hip_runtime.h"
#include "includes.h"




hipError_t sortWithCuda(int *a, size_t size, float* time);

typedef long long int64;
typedef unsigned long long uint64;
__global__ void swapOnKernel(int *a, int size)
{
int i = blockDim.x * blockIdx.x + threadIdx.x * 2;
int cacheFirst;
int cacheSecond;
int cacheThird;

for (int j = 0; j < size/2 + 1; j++) {

if(i+1 < size) {
cacheFirst = a[i];
cacheSecond = a[i+1];

if(cacheFirst > cacheSecond) {
int temp = cacheFirst;
a[i] = cacheSecond;
cacheSecond = a[i+1] = temp;
}
}

if(i+2 < size) {
cacheThird = a[i+2];
if(cacheSecond > cacheThird) {
int temp = cacheSecond;
a[i+1] = cacheThird;
a[i+2] = temp;
}
}

__syncthreads();
}

}