#include "hip/hip_runtime.h"
#include "includes.h"
#define B 2


/*
*/

__global__ void cudaAcc_GetPowerSpectrum_kernel2( int NumDataPoints, float2* FreqData, float* PowerSpectrum)
{
const int i = blockIdx.x * blockDim.x*B + threadIdx.x;

float ax[B];
float ay[B];

#pragma unroll
for (int k=0;k<B;k++)
{
ax[k] = FreqData[i+k*blockDim.x].x;
ay[k] = FreqData[i+k*blockDim.x].y;
}
//		PowerSpectrum[i] = freqData.x * freqData.x + freqData.y * freqData.y;

#pragma unroll
for (int k=0;k<B;k++)
{
PowerSpectrum[i+k*blockDim.x] = __fadd_rn( __fmul_rn(ax[k],ax[k]),__fmul_rn(ay[k],ay[k]));
}
}