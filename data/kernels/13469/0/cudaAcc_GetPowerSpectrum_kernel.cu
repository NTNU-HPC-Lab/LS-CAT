#include "hip/hip_runtime.h"
#include "includes.h"
#define B 2


/*
*/

__global__ void cudaAcc_GetPowerSpectrum_kernel( int NumDataPoints, float2* FreqData, float* PowerSpectrum) {
const int i = blockIdx.x * blockDim.x + threadIdx.x;

//	if (i < NumDataPoints) {
float ax = FreqData[i].x;
float ay = FreqData[i].y;
//		PowerSpectrum[i] = freqData.x * freqData.x + freqData.y * freqData.y;
PowerSpectrum[i] = __fadd_rn( __fmul_rn(ax,ax),__fmul_rn(ay,ay));
//	}
}