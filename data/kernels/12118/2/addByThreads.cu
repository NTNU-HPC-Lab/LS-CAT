#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addByThreads(int *a, int *b, int *c)
{
// a block can be split into parallel threads.
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}