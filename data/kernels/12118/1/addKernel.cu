#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *a, int *b, int *c)
{
// each parallel invocation of add() is referred to as a block.
// The set of blocks is referred to as a grid.
// Each invocation can refer to its block index using blockIdx.x.
// By using blockIdx.x to index into the array, each block handles a different index.
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}