#include "hip/hip_runtime.h"
#include "includes.h"



#define N (2048*2048)
#define N_THREADS_PER_BLOCK 512

// Adapt vector addition to use both blocks and threads

__global__ void addByCombine(int *a, int *b, int *c)
{
// use the built-in variable blockDim.x for threads per block
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}