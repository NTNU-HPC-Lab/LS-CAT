#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ecuaciones(int a, int b, int c, float *sol){

int index = threadIdx.x;
float d = 0;
float x=0, y=0;
d = b*b-4*a*c;
if (d > 0) {
x = (-b+sqrt(d))/(2*a);
y = (-b-sqrt(d))/(2*a);
sol[index] = x;
sol[index+1]=y;
}
else if (d == 0) {
x = (-b)/(2*a);
sol[index] = x;
}
}