#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumaVector(float *v1, float *v2, float *res){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index < TAM_V)
res[index] = v1[index] + v2[index];

}