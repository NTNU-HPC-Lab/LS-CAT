#include "hip/hip_runtime.h"
#include "includes.h"

#define N 24


__global__ void muestraIndice(float *a, float *b, float *c){

int global = blockIdx.x * blockDim.x + threadIdx.x;

if(global < N){
a[global] = threadIdx.x;
b[global] = blockIdx.x;
c[global] = global;
}

}