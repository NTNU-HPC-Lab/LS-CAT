#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceVector(float *v1, float *v2, float *res){


int index = blockIdx.x * blockDim.x + threadIdx.x;
int index2;

for (int i = blockDim.x/2; i>=1; i=i/2){

if(threadIdx.x < i){
index2 = index + i;
v1[index] += v1[index2];
}
__syncthreads();

}
if(threadIdx.x==0)
res[blockIdx.x] = v1[index];

}