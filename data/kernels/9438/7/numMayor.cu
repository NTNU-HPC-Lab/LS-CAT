#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void numMayor(float *d_v, float *d_pos){


float temp = 0,pos=0;
for(int i=threadIdx.x; i<blockDim.x;i++){
if(d_v[i] > temp){
temp = d_v[i];
pos = i;
}

}
__syncthreads();
if(pos>d_pos[threadIdx.x])
d_pos[threadIdx.x] = pos;
d_v[threadIdx.x] = temp;

}