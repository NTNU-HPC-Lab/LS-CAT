#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumaVectores (float * d_a, float *d_b, float * d_c) {

int index = blockIdx.x*blockDim.x+threadIdx.x;
if (index < N )
d_c[index] = d_a[index] +d_b[index];
}