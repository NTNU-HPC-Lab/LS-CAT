#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void GaussianNBVarKernel(const float *d_data, const int *d_labels, const float *feature_means_, float *feature_vars_, const int *class_count_, const unsigned int n_samples_, const unsigned int n_classes_, const unsigned int n_features_) {

// Each thread will take care of one feature for all training samples
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0, row = 0;

// Calculate variances
if (feat_col < n_features_) {        /* End condition check */
for (i = 0; i < n_samples_; ++i) { /* For each sample */
row = d_labels[i];
feature_vars_[RM_Index(row, feat_col, n_features_)] +=
pow(d_data[RM_Index(i, feat_col, n_features_)] -
feature_means_[RM_Index(row, feat_col, n_features_)],
2);
}

// Calculate coefficients
for (i = 0; i < n_classes_; ++i) { /* For each class */
feature_vars_[RM_Index(i, feat_col, n_features_)] /= class_count_[i];
}
}
}