#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void ComplementNBCalcKernel(const float *d_data, const int *d_labels, float *per_class_feature_sum_, float *per_feature_sum_, unsigned int n_samples_, unsigned int n_features_) {

// Each thread will take care of one term for all docs
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0, row = 0;

if (feat_col < n_features_) { // End condition check
// For each document / sample
for (i = 0; i < n_samples_; ++i) {
row = d_labels[i];

// No race condition since each thread deals with one feature only
// This is embarrasingly parallel
per_class_feature_sum_[RM_Index(row, feat_col, n_features_)] +=
d_data[RM_Index(i, feat_col, n_features_)];

per_feature_sum_[feat_col] += d_data[RM_Index(i, feat_col, n_features_)];
}
}
return;
}