#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void ComplementNBNormalizeKernel(float *feature_weights_, float *per_class_sum_, unsigned int n_classes_, unsigned int n_features_) {
// Each thread will take one feature
int feat_col = threadIdx.x + (blockIdx.x * blockDim.x);
unsigned int i = 0;

if (feat_col < n_features_) {        /* Boundary condition check */
for (i = 0; i < n_classes_; ++i) { // For each class
feature_weights_[RM_Index(i, feat_col, n_features_)] /= per_class_sum_[i];
}
}
}