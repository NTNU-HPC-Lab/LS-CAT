#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void ComplementNBLearnKernel(float *feature_weights_, float *per_class_feature_sum_, float *per_feature_sum_, float *per_class_sum_, float all_sum_, unsigned int n_classes_, unsigned int n_features_) {
// Each thread will take one feature
unsigned int tidx = threadIdx.x;
int feat_col = tidx + (blockIdx.x * blockDim.x);

unsigned int i = 0;
float den_sum = 0;
float num_sum = 0;

if (feat_col < n_features_) {        /* Boundary check */
for (i = 0; i < n_classes_; ++i) { /* For each class */
den_sum = all_sum_ - per_class_sum_[i];
num_sum = per_feature_sum_[feat_col] -
per_class_feature_sum_[RM_Index(i, feat_col, n_features_)];

feature_weights_[RM_Index(i, feat_col, n_features_)] =
log(num_sum + 1.0) - log(den_sum + n_features_);
}
}
}