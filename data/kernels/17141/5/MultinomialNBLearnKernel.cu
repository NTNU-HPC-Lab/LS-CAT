#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void MultinomialNBLearnKernel(float *feature_probs, float *class_priors, const float *d_row_sums, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

/* Each thread will take one term */
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0;

if (feat_col < n_features_) { /* End condition check */
/* For each label */
for (i = 0; i < n_classes_; ++i) {
// Alpha is default set to 1.0 for laplacian smoothing
feature_probs[RM_Index(i, feat_col, n_features_)] =
log((feature_probs[RM_Index(i, feat_col, n_features_)] + 1.0) /
(d_row_sums[i] - (n_features_ * 1.0)));

if (feat_col == 0) {
class_priors[i] = log(class_priors[i] / (float)n_samples_);
}
}
}
}