#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void BernoulliNBLearnKernel(float *feature_probs, float *class_count_, const float *d_row_sums, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

// Each thread will take one term
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0;

if (feat_col < n_features_) { // End condition check
// For each label
for (i = 0; i < n_classes_; ++i) {
feature_probs[RM_Index(i, feat_col, n_features_)] /=
class_count_[i]; // d_row_sums[i];

if (feat_col == 0) {
class_count_[i] = class_count_[i] / (float)n_samples_;
}
}
}
}