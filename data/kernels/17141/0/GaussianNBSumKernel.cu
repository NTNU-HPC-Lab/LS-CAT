#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void GaussianNBSumKernel(const float *d_data, const int *d_labels, float *feature_means_, int *class_count_, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

// Each thread will take care of one feature for all training samples
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0, row = 0;

if (feat_col < n_features_) { /* End condition check */

for (i = 0; i < n_samples_; ++i) { /* For each training sample */
row = d_labels[i];

// No race condition since each thread deals with one feature only
feature_means_[RM_Index(row, feat_col, n_features_)] +=
d_data[RM_Index(i, feat_col, n_features_)];

// WARNING: thread divergence :/
if (feat_col == 0) {
class_count_[row] += 1;
}
}
}
return;
}