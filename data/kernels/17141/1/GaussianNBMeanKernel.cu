#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void GaussianNBMeanKernel(float *feature_means_, int *class_count_, float *class_priors_, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

// Each thread will take care of one feature for all training samples
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0;

if (feat_col < n_features_) { /* End condition check */

/* Calculate Means */
for (i = 0; i < n_classes_; ++i) { /* For each class */
feature_means_[RM_Index(i, feat_col, n_features_)] /= class_count_[i];

// WARNING: thread divergence
// Calculating Class priors
if (feat_col == 0) {
class_priors_[i] = (float)class_count_[i] / n_samples_;
}
}
}
}