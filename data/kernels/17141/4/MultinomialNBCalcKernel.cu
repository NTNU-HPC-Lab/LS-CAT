#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
return (row * width + col);
}
__global__ void MultinomialNBCalcKernel(const float *d_data, const int *d_labels, float *feature_probs, float *class_priors, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

// Each thread will take care of one term for all docs
unsigned int tidx = threadIdx.x;
unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
unsigned int i = 0, row = 0;

if (feat_col < n_features_) { /* End condition check */

/* For each document / sample */
for (i = 0; i < n_samples_; ++i) {
row = d_labels[i];

// No race condition since each thread deals with one feature only
feature_probs[RM_Index(row, feat_col, n_features_)] +=
d_data[RM_Index(i, feat_col, n_features_)];

// WARNING: thread divergence :(
if (feat_col == 0) {
class_priors[row] += 1;
}
}
}
return;
}