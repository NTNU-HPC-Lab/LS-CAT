#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_divide(float * dst, float *numerator, float *denominator, int width, int height)
{
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;

if(row < height && col < width)
{
int index = row * width + col;
if(denominator[index] > 0.0000001)
{
dst[index] = numerator[index] / denominator[index];
}
else
{
dst[index] = 0;
}
//        printf("dst[%d] = %f\n", index, dst[index]);
}
}