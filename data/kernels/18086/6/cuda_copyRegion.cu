#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_copyRegion(unsigned char *dst, unsigned char *src,int stepDst, int stepSrc, int dst_width, int dst_height, int src_width, int src_height, int dst_xoffset, int dst_yoffset, int dst_widthToCrop, int dst_heightToCrop, int src_xoffset, int src_yoffset, int src_widthToCrop, int src_heightToCrop, int numChannel)
{
//    printf("stepSrc - Dst = %d - %d\n", stepSrc, stepDst);
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;

int dst_col = col + dst_xoffset;
int dst_row = row + dst_yoffset;

int src_col = col + src_xoffset;
int src_row = row + src_yoffset;


if(row < dst_heightToCrop && col < dst_widthToCrop && dst_col < dst_width&& dst_row < dst_height)
{
if(numChannel==1)
{
dst[dst_row * (stepDst) + dst_col] = src[src_row * (stepSrc) + src_col];
}
if(numChannel==3)
{
int dst_step = dst_row * (stepDst) + dst_col;
int src_step = src_row * (stepSrc) + src_col;
dst[3 * dst_step] = src[3 * src_step];
dst[3 * dst_step + 1] = src[3 * src_step + 1];
dst[3 * dst_step + 2] = src[3 * src_step + 2];
}
}
}