#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cuda_copyRegion.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned char *dst = NULL;
hipMalloc(&dst, XSIZE*YSIZE);
unsigned char *src = NULL;
hipMalloc(&src, XSIZE*YSIZE);
int stepDst = 1;
int stepSrc = 1;
int dst_width = XSIZE;
int dst_height = YSIZE;
int src_width = XSIZE;
int src_height = YSIZE;
int dst_xoffset = 1;
int dst_yoffset = 1;
int dst_widthToCrop = XSIZE;
int dst_heightToCrop = YSIZE;
int src_xoffset = 1;
int src_yoffset = 1;
int src_widthToCrop = XSIZE;
int src_heightToCrop = YSIZE;
int numChannel = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
cuda_copyRegion<<<gridBlock,threadBlock>>>(dst,src,stepDst,stepSrc,dst_width,dst_height,src_width,src_height,dst_xoffset,dst_yoffset,dst_widthToCrop,dst_heightToCrop,src_xoffset,src_yoffset,src_widthToCrop,src_heightToCrop,numChannel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
cuda_copyRegion<<<gridBlock,threadBlock>>>(dst,src,stepDst,stepSrc,dst_width,dst_height,src_width,src_height,dst_xoffset,dst_yoffset,dst_widthToCrop,dst_heightToCrop,src_xoffset,src_yoffset,src_widthToCrop,src_heightToCrop,numChannel);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
cuda_copyRegion<<<gridBlock,threadBlock>>>(dst,src,stepDst,stepSrc,dst_width,dst_height,src_width,src_height,dst_xoffset,dst_yoffset,dst_widthToCrop,dst_heightToCrop,src_xoffset,src_yoffset,src_widthToCrop,src_heightToCrop,numChannel);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}