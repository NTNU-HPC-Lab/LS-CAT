#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_multiply(float *dst, float *src1, float *src2, int width, int height)
{
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;

if(row < height && col < width)
{
int index = row * width + col;
dst[index] = src1[index] * src2[index];
#ifdef debug
printf("multiply dst[%d] = %f\n", index, dst[index]);
#endif
}
}