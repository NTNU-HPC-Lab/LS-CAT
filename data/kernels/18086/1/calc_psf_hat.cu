#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_psf_hat(float* d_psf, float *d_psf_hat, int psf_rows, int psf_cols)
{
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;
if(row < psf_rows && col < psf_cols)
{
int index = (psf_rows - row - 1) * psf_cols + psf_cols - col - 1;
d_psf_hat[index] = d_psf[row * psf_cols + col];
//        if(d_psf_hat[index] > 0)
//            printf("psf_hat[%d] = %f - psf[%d] = %f\n", index,d_psf_hat[index], row * psf_cols + col, d_psf[row * psf_cols + col]);
}
}