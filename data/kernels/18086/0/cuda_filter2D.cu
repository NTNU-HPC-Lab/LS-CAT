#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_filter2D(float *dst, float *src, float *kernel, int src_width, int src_height, int kernel_rows, int kernel_cols)
{
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;
if(row < src_height && col < src_width)
{
float sum = 0;
for(int i = 0; i < kernel_rows; i++)
{
for(int j = 0; j < kernel_cols; j++)
{
if(row + i - (kernel_rows - 1) / 2 >= 0 &&
col + j - (kernel_cols - 1) / 2 >= 0 &&
col + j - (kernel_cols - 1) / 2 < src_width &&
row + i - (kernel_rows - 1) / 2 < src_height)
{
sum = sum + kernel[i * kernel_cols + j] * (float)src[(row + i - (kernel_rows - 1)/2) * src_width + col +j - (kernel_rows - 1)/2];
}
}
}
dst[row *src_width + col] = (sum <0)?0:(sum>255?255:float(sum));
#ifdef debug
printf("filter2D: dst[%d] = %f\n", row * src_width + col, dst[row * src_width + col]);
#endif
}
}