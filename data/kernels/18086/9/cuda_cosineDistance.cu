#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_cosineDistance(double *x, double* y, int64_t len, double *dot_product, double *norm_x, double*norm_y)
{
int64_t idx = threadIdx.x + blockIdx.x * blockDim.x;

int64_t cacheIdx = threadIdx.x;

__shared__ double dot_cache[threadsPerBlock];
__shared__ double norm_x_cache[threadsPerBlock];
__shared__ double norm_y_cache[threadsPerBlock];

double dot_tmp = 0;
double norm_x_tmp = 0;
double norm_y_tmp = 0;

while(idx < len)
{
dot_tmp += x[idx] * y[idx];
norm_x_tmp += x[idx] * x[idx];
norm_y_tmp += y[idx] * y[idx];
idx += blockDim.x * gridDim.x;
}
dot_cache[cacheIdx] = dot_tmp;
norm_x_cache[cacheIdx] = norm_x_tmp;
norm_y_cache[cacheIdx] = norm_y_tmp;
__syncthreads();

int64_t i = blockDim.x/2;
while(i!=0)
{
if(cacheIdx < i)
{
dot_cache[cacheIdx] += dot_cache[cacheIdx + i];
norm_x_cache[cacheIdx] += norm_x_cache[cacheIdx + i];
norm_y_cache[cacheIdx] += norm_y_cache[cacheIdx + i];
}
__syncthreads();
i/=2;
}

if(cacheIdx == 0)
{
dot_product[blockIdx.x] = dot_cache[0];
norm_x[blockIdx.x] = norm_x_cache[0];
norm_y[blockIdx.x] = norm_y_cache[0];
}
}