#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void segmentSieve(char *primes, uint64_t max) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if(index>0){
const uint64_t maxRoot = sqrt((double)max);
int low = maxRoot*index;
int high = low + maxRoot;
if(high > max) high = max;
for (int i = 2; i < maxRoot; i++){ //sqrt(n)lglg(sqrt(n))
if(primes[i]==0){
int loLim = (low / i) * i;
if (loLim < low)
loLim += i;
for (int j=loLim; j<high; j+=i)
primes[j] = 1;
}

}
}
}