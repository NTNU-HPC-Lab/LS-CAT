#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void sieveOfEratosthenes(char *primes, uint64_t max) {
primes[0] = 1; // value of 1 means the number is NOT prime
primes[1] = 1; // numbers "0" and "1" are not prime numbers
int index = blockIdx.x * blockDim.x + threadIdx.x;
const uint64_t maxRoot = sqrt((double)max);
// make sure index won't go out of bounds,
// also don't execute it on index 1
if (index <= maxRoot && primes[index] == 0 && index > 1 ){
// mark off the composite numbers
for (int j = index * index; j < max; j += index){
primes[j] = 1;
}

}
}