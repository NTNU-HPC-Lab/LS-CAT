#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(bool *B,int *number,int order){
int num = 0;
int idx = blockIdx.x * blockDim.x + threadIdx.x;

//if(idx==0) printf("ORDER%d\n",order);
//printf("IDX%d\n",idx);
if(idx<order){

for(int i = 0 ; i<order; i++)
if(B[idx*order+i]==1)
num ++; //= B[idx*order + i];
//if(B[idx]==1)
//printf("CUDANUM%d\n",num);
number[idx] = num;
//atomicAdd(number,num);
//printf("NUMBER%d\n",number);//<<endl;
}
}