#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void logicalkernel(bool *A, bool *B, int *neighbours, int order ,int degree) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx <order){
for(int i=0 ; i <  degree ; i++){
int n = neighbours[idx*degree + i ];
for(int j = 0; j < order; j++){
B[idx * order+ j] = B[idx*order+j] || A[n*order+j];
}
}
}
}