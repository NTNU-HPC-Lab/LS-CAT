#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bigstencil(int* in, int* out) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
out[i] = in[i] + 2;
}