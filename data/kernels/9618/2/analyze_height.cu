#include "hip/hip_runtime.h"
#include "includes.h"
/*
cudaStructTest
testing/optimizing how to access/manipulate/return
structures in cuda.
*/



#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000
/*#define BLOCKS 2
#define THREADS 5*/

int cuda_setup(int computeCapability);

typedef struct{
int id;
int age;
int height;
} Person;


// Declare the Cuda kernels and any Cuda functions




__global__ void analyze_height(Person *people, int *statResults)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;

if(id < N)
{
Person person = people[id];

if(person.height != 6)
{
statResults[id] = 1;
}
else
{
statResults[id] = 0;
}
}

}