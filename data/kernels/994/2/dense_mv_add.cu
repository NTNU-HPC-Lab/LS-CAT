#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dense_mv_add(size_t sz, float_t* src, float_t* dest)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
if(index < sz)
{
dest[index] += src[index];
}
}