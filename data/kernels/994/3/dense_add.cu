#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dense_add(size_t sz, float_t* src, float_t* dest)
{
size_t srcIndex = threadIdx.x;
size_t destIndex = blockIdx.x*blockDim.x + threadIdx.x;
if(destIndex < sz)
{
dest[destIndex] += src[srcIndex];
}
}