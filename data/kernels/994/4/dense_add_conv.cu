#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dense_add_conv(size_t sz, float_t* src, float_t* dest, size_t bias_dim)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
// size_t src_index = index%bias_dim;
if(index < sz)
{
dest[index] += src[threadIdx.x];
}
}