#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void skip_add(size_t sz, float_t* f1, float* skip_out_sum)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

if(index < sz)
{
skip_out_sum[index] += f1[index];
}
}