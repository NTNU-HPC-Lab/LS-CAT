#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose(size_t sz, float_t* src, float_t* dest, size_t src_width, size_t src_height)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

size_t i = index/src_width ;
size_t j = index%src_width;

size_t dest_index = j*src_height+i;
if(index < sz)
{
dest[dest_index] = src[index];
}
}