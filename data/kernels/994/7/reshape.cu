#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reshape(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
if(index < num_values)
{
size_t src_index = (index/ld_dest)*ld_src+ index%ld_dest;
dest[index] = src[src_index];
}
}