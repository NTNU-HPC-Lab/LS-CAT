#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(size_t sz, float_t* src, float_t* dest)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

if(index < sz)
{
dest[index]=src[index];
}
}