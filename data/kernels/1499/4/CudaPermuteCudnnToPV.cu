#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaPermuteCudnnToPV( float *dest, float *src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY) {
// parameter dimensions are in dest PV format
int srcNx         = nx / manyScaleX;
int srcNy         = ny / manyScaleY;
int srcInFeatures = inFeatures * manyScaleX * manyScaleY;

int kDest = (blockIdx.x * blockDim.x) + threadIdx.x;
if (kDest < outFeatures * ny * nx * inFeatures) {
int kOF = kDest / (ny * nx * inFeatures);
int kY  = (kDest % (ny * nx * inFeatures)) / (nx * inFeatures);
int kX  = (kDest % (nx * inFeatures)) / inFeatures;
int kIF = (kDest % inFeatures);

// Recalculate x, y, and f based on manyScale
kIF = kIF + inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
kX  = kX / manyScaleX;
kY  = kY / manyScaleY;

int sOF = srcInFeatures * srcNy * srcNx;
int sIF = srcNy * srcNx;
int sY  = srcNx;

int kSrc = kOF * sOF + kIF * sIF + kY * sY + kX;

dest[kDest] = src[kSrc];
}
}