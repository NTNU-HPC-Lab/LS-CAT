#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaPermutePVToCudnn( float *dest, float *src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY, int cropX, int cropY) {
// parameter dimensions are in source PV format
int destNx         = (nx - 2 * cropX) / manyScaleX;
int destNy         = (ny - 2 * cropY) / manyScaleY;
int destInFeatures = inFeatures * manyScaleX * manyScaleY;

int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
if (kSrc < outFeatures * ny * nx * inFeatures) {
int kOF = kSrc / (ny * nx * inFeatures);
int kY  = (kSrc % (ny * nx * inFeatures)) / (nx * inFeatures);
int kX  = (kSrc % (nx * inFeatures)) / inFeatures;
int kIF = (kSrc % inFeatures);

// check if in bounds
if (kX < cropX || kX >= nx - cropX) {
return;
}
else {
kX = kX - cropX;
}
if (kY < cropY || kY >= ny - cropY) {
return;
}
else {
kY = kY - cropY;
}

// Recalculate x, y, and f based on manyScale
kIF = kIF + inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
kX  = kX / manyScaleX;
kY  = kY / manyScaleY;

int sOF = destInFeatures * destNy * destNx;
int sIF = destNy * destNx;
int sY  = destNx;

int kDest = kOF * sOF + kIF * sIF + kY * sY + kX;

dest[kDest] = src[kSrc];
}
}