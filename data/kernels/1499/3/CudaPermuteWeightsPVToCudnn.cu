#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaPermuteWeightsPVToCudnn( float *dest, float *src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY) {
// Parameter dimensions are PV source dimensions
int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
if (kSrc < outFeatures * manyScaleX * manyScaleY * ny * nx * inFeatures) {
int kOF = kSrc / (ny * nx * inFeatures);
int kY  = (kSrc % (ny * nx * inFeatures)) / (nx * inFeatures);
int kX  = (kSrc % (nx * inFeatures)) / inFeatures;
int kIF = (kSrc % inFeatures);

int sOF = inFeatures * ny * nx;
int sIF = ny * nx;
int sY  = nx;

int kDest = kOF * sOF + kIF * sIF + (ny - kY - 1) * sY + (nx - kX - 1);

dest[kDest] = src[kSrc];
}
}