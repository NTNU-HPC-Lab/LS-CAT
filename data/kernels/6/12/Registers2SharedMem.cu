#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Registers2SharedMem(float * outFloat, int iSize)
{
/* Amount of shared memory is determined by host call */
extern __shared__ float s_memoryA[];
/* Variable in register */
float r_var;
/* Generate global index */
int iID = blockDim.x * blockIdx.x + threadIdx.x;
/* Get the number of available threads */
int iNumThreads = blockDim.x * gridDim.x;
/* Calculate number of elements */
int iNumElements = iSize / sizeof(float);
/* Read global memory (coalesce) to shared memory */
for(int i = iID; i < iNumElements; i += iNumThreads)
s_memoryA[i] = r_var;
/* Conditionally assign register var, so it won't get optimized */
if(iID == 0) outFloat[0] = r_var;
}