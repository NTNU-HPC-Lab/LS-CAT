#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SharedMem2Registers(float * outFloat, int iSize)
{
/* Amount of shared memory is determined by host call */
extern __shared__ float s_memoryA[];
/* Variable in register */
float r_var;
/* Generate global index */
int iID = blockDim.x * blockIdx.x + threadIdx.x;
/* Get the number of available threads */
int iNumThreads = blockDim.x * gridDim.x;
/* Calculate number of elements */
int iNumElements = iSize / sizeof(float);
/* Read global memory (coalesce) to shared memory */
for(int i = iID; i < iNumElements; i += iNumThreads)
r_var = s_memoryA[i];
/* Conditionally assign register var, so it won't get optimized */
if(iID == 0) outFloat[0] = r_var;
}