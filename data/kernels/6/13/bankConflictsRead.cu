#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bankConflictsRead(float *outFloat, int iStride, unsigned long long *ullTime)
{
/* Static size of shared memory */
__shared__ float s_memoryA[2024];
/* Variable in register */
float r_var;
/* Start measure clock cycles */
unsigned long long startTime = clock64();
/* Access data from shared memory to register */
r_var = s_memoryA[threadIdx.x*iStride];
/* End measure clock cycles */
*ullTime = clock64() - startTime;
/* Conditionally assign register var, so it won't get optimized */
if(threadIdx.x == 0) outFloat[0] = r_var;
}