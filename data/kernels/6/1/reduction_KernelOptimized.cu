#include "hip/hip_runtime.h"
#include "includes.h"
/**************************************************************************************************
*
*       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
*
*                 Gruppe : gpucomp02
*
*                   File : main.cu
*
*                Purpose : Reduction
*
**************************************************************************************************/


const static int DEFAULT_MATRIX_SIZE = 1024;
const static int DEFAULT_BLOCK_DIM   =  128;

//
// Function Prototypes
//
void printHelp(char *);

//
// CPU
//
__global__ void reduction_KernelOptimized(int numElements, float* dataIn, float* dataOut)
{
extern __shared__ float sPartArray[];

const int tid = threadIdx.x;
unsigned int elementId = blockIdx.x * blockDim.x + threadIdx.x;

sPartArray[tid] = dataIn[elementId];
__syncthreads();

for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
if(tid < s) {
sPartArray[tid] += sPartArray[tid + s];
}
__syncthreads();
}

if (tid == 0) {
dataOut[blockIdx.x] = sPartArray[0];
}
}