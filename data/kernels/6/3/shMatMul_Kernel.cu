#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shMatMul_Kernel(int matrixSize, float* matrixA, float* matrixB, float* matrixC)
{
extern __shared__ float sh_Mem[];
int tilewidth = blockDim.x;
float *sh_MatrixA = &(sh_Mem[0]);
float *sh_MatrixB = &(sh_Mem[1*tilewidth*tilewidth]);
//float *sh_MatrixC= &(sh_Mem[2*tilewidth*tilewidth]);

int elementIdx = blockIdx.x * blockDim.x + threadIdx.x; // Col
int elementIdy = blockIdx.y * blockDim.y + threadIdx.y; // Row

int elementId = elementIdy * matrixSize + elementIdx;
float CValue = 0;
if (elementIdx < matrixSize && elementIdy < matrixSize) {
for(int m=0; m < (matrixSize/tilewidth); ++m)
{
sh_MatrixA[tilewidth*threadIdx.y + threadIdx.x] = matrixA[elementIdy*matrixSize + (m*tilewidth+threadIdx.x)];
sh_MatrixB[tilewidth*threadIdx.y + threadIdx.x] = matrixB[elementIdx + (m*tilewidth+threadIdx.y)*matrixSize];
__syncthreads();

for(int k=0; k<tilewidth; ++k)
CValue += sh_MatrixA[tilewidth*threadIdx.y + k] * sh_MatrixB[tilewidth*k + threadIdx.x];
__syncthreads();
}
matrixC[elementId] = CValue;
}
}