#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updatePosition_Kernel(int numElements, float4* bodyPos, float3* bodySpeed)
{
int elementId = blockIdx.x * blockDim.x + threadIdx.x;

float4 elementPosMass;
float3 elementSpeed;

if (elementId < numElements) {
elementPosMass = bodyPos[elementId];
elementSpeed = bodySpeed[elementId];

elementPosMass.x += elementSpeed.x * TIMESTEP;
elementPosMass.y += elementSpeed.y * TIMESTEP;
elementPosMass.z += elementSpeed.z * TIMESTEP;

bodyPos[elementId] = elementPosMass;
}
}