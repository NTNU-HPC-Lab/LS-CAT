#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SharedMem2globalMem(float * d_memoryA, int iSize)
{
/* Amount of shared memory is determined by host call */
extern __shared__ float s_memoryA[];
/* Generate global index */
int iID = blockDim.x * blockIdx.x + threadIdx.x;
/* Get the number of available threads */
int iNumThreads = blockDim.x * gridDim.x;
/* Calculate number of elements */
int iNumElements = iSize / sizeof(float);
/* Read global memory (coalesce) to shared memory */
for(int i = iID; i < iNumElements; i += iNumThreads)
d_memoryA[i] = s_memoryA[i];
}