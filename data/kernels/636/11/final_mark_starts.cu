#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void final_mark_starts( uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t r, uint32_t hash_count)
{
uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
if(t_index < hash_count) {
uint32_t t_prev_index = (t_index-1) % hash_count; // wrap around at index 0

uint32_t index = sort_indices[t_index];
uint32_t prev_index = sort_indices[t_prev_index];

unsigned char* hash = hashes+index*30*sizeof(unsigned char)+r*3;
unsigned char* prev_hash = hashes+prev_index*30*sizeof(unsigned char)+r*3;

uint64_t key = ((uint64_t)hash[0]) << 40 | ((uint64_t)hash[1]) << 32 | hash[2] << 24;
key |= hash[3] << 16 | hash[4] << 8 | hash[5];

uint64_t prev_key = ((uint64_t)prev_hash[0]) << 40 | ((uint64_t)prev_hash[1]) << 32 | prev_hash[2] << 24;
prev_key |= prev_hash[3] << 16 | prev_hash[4] << 8 | prev_hash[5];

if((key ^ prev_key) != 0) {
off_map[t_index] = 1;
}
}
}