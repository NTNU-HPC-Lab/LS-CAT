#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void final_map_index_to_prefix( uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
if(t_index < size) {
uint32_t index = sort_indices[t_index];
unsigned char* hash = hashes+index*30*sizeof(unsigned char)+r*3;

uint64_t key = ((uint64_t)hash[0]) << 40 | ((uint64_t)hash[1]) << 32 | hash[2] << 24;
key |= hash[3] << 16 | hash[4] << 8 | hash[5];

uint32_t lookup_idx = off_map[t_index];

uint64_t count = comb_count[lookup_idx];
uint64_t sum = comb_sum[lookup_idx];
for(int i=(sum-count); i<sum; i++) {
comb_prefix[i] = lookup_idx;
}
}
}