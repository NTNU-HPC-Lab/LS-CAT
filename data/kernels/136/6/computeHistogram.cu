#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeHistogram(unsigned int  *buffer, int size, unsigned int *histo )
{
__shared__ unsigned int temp[1024];

temp[threadIdx.x + 0] = 0;
temp[threadIdx.x + 256] = 0;
temp[threadIdx.x + 512] = 0;
temp[threadIdx.x + 768] = 0;
__syncthreads();

int i = threadIdx.x + blockIdx.x * blockDim.x;
int offset = blockDim.x * gridDim.x;
while (i < size)
{
atomicAdd( &temp[buffer[i]], 1);
i += offset;
}
__syncthreads();


atomicAdd( &(histo[threadIdx.x + 0]), temp[threadIdx.x + 0] );
atomicAdd( &(histo[threadIdx.x + 256]), temp[threadIdx.x + 256] );
atomicAdd( &(histo[threadIdx.x + 512]), temp[threadIdx.x + 512] );
atomicAdd( &(histo[threadIdx.x + 768]), temp[threadIdx.x + 768] );

}