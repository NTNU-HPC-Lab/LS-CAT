#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float f(float x)
{
return 4.f / (1.f + x * x);
}
__global__ void multMatrixGPU(const float *matrixA, const size_t rowA, const size_t columnA, const float *matrixB, const size_t rowB, const size_t columnB, float *matrixC)
{
__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
__shared__ float ds_B[TILE_SIZE][TILE_SIZE];

int tx = threadIdx.x;
int ty = threadIdx.y;

int row = blockIdx.y * blockDim.y + ty;
int col = blockIdx.x * blockDim.x + tx;

float cValue = 0.0;

for (int t = 0; t < (columnA - 1) / TILE_SIZE + 1; t++)
{
if (t * TILE_SIZE + tx < columnA && row < rowA)
{
ds_A[ty][tx] = matrixA[row * columnA + t * TILE_SIZE + tx];
} else {
ds_A[ty][tx] = 0.0;
}

if (t * TILE_SIZE + ty < rowB && col < columnB)
{
ds_B[ty][tx] = matrixB[(t * TILE_SIZE + ty) * columnB + col];
} else {
ds_B[ty][tx] = 0.0;
}

__syncthreads();

for (int i = 0; i < TILE_SIZE; i++) {
cValue += ds_A[ty][i] * ds_B[i][tx];
}

__syncthreads();
}

if (row < rowA && col < columnB) {
matrixC[row * columnB + col] = cValue;
}
}