#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float f(float x)
{
return 4.f / (1.f + x * x);
}
__global__ void searchGPU(float *data, const float x, int *result)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (data[idx] == x)
{
result[0] = static_cast<int>(data[idx]);
result[1] = idx;
}
}