#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE 16


__device__ float f(float x)
{
return 4.f / (1.f + x * x);
}
__global__ void transGPU(const float *inMatrix, float *outMatrix, const size_t row, const size_t column)
{
size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

if ((xIndex < column) && (yIndex < row))
{
size_t inIndex = yIndex * column + xIndex;
size_t outIndex = xIndex * row + yIndex;

outMatrix[outIndex] = inMatrix[inIndex];
}
}