#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float f(float x)
{
return 4.f / (1.f + x * x);
}
__global__ void calcIntegralGPU(float *array, const float h, const long stepCount, const int threads, const int blocks)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = idx; i < stepCount; i+= threads * blocks)
{
float x = (i + 0.5f) * h;
array[idx] += f(x);
}
}