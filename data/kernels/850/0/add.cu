#include "hip/hip_runtime.h"
#include "includes.h"
/**
*	This is my first program in learning parallel programming using CUDA.
*	Equivalent to a hello World program :-)
*	This program basically performs two tasks:
*	1. It selects suitable CUDA enabled device(GPU) and prints the device properties
*	2. It demonstrate basic parallel addition of two arrays on the device(GPU) using add kernel.
*	Author: Shubham Singh
**/


#define N 10						/*N is size of arrays*/

using namespace std;

/************************************************************************************************************
*	Function:	Kernel to perform addition of two arrays in parallel on device(GPU)
*	Input:		Takes 3 pointer to int variables pointing to some memory locations on the device(GPU)
*	Output:		None
************************************************************************************************************/

__global__ void add(int *a, int *b, int *c)
{
int i = blockIdx.x;				/*blockIDx.x holds ID of block and acts as index*/
if (i < N)
c[i] = a[i] + b[i];
}