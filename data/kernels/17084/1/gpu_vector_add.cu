#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_vector_add(float *out, float *a, float *b, int n) {
// built-in variable blockDim.x describes amount threads per block

int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < n)
out[tid] = a[tid] + b[tid];


// more advanced version - handling arbitrary vector/kernel size
// int i = blockIdx.x * blockDim.x + threadIdx.x;
// int step = gridDim.x * blockDim.x;

// for(; i < n; i += step){
//     out[i] = a[i] + b[i];
// }
}