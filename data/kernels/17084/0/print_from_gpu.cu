#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_from_gpu(void) {
int tidx = blockIdx.x*blockDim.x+threadIdx.x;
printf("Hello from device! My threadId = blockIdx.x *blockDim.x + threadIdx.x <=> %d = %d * %d + %d \n",
tidx, blockIdx.x, blockDim.x, threadIdx.x);
}