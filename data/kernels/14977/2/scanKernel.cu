#include "hip/hip_runtime.h"
#include "includes.h"

// Number of elements to put in the test array
#define TEST_SIZE 16
#define NUM_BINS 10

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 4
// Functions to reduce with
#define ADD 0
#define MIN 1
#define MAX 2
// Device functions

__global__ void scanKernel(unsigned int* d_cdf, unsigned int* d_input, const size_t array_size)
{
__shared__ unsigned int temp[BLOCK_WIDTH<<1];
int bx = blockIdx.x;
int tx = threadIdx.x;
int index = BLOCK_WIDTH * bx + tx;
int offset = 1;

if(2*index + 1 < array_size) {
temp[2*index] = d_input[2*index];
temp[2*index + 1] = d_input[2*index + 1];
}

// Up-sweep
for(int powOf2 = (2*BLOCK_WIDTH)>>1; powOf2 > 0; powOf2 >>= 1) {
__syncthreads();
if(tx < powOf2) {
int idx1 = offset*(2*tx + 1) - 1 + 2*BLOCK_WIDTH*bx;
int idx2 = offset*(2*tx + 2) - 1 + 2*BLOCK_WIDTH*bx;
temp[idx2] += temp[idx1];
}
offset <<= 1;
}

__syncthreads();
///// The below will need to be remembered for multiple blocks /////
if(tx == 0) {
temp[2*BLOCK_WIDTH*(bx + 1) - 1] = 0;
}

// Down-sweep
for(int powOf2 = 1; powOf2 < 2*BLOCK_WIDTH; powOf2 <<= 1) {
offset >>= 1;
__syncthreads();
if(tx < powOf2) {
int idx1 = offset*(2*tx + 1) - 1 + 2*BLOCK_WIDTH*bx;
int idx2 = offset*(2*tx + 2) - 1 + 2*BLOCK_WIDTH*bx;
unsigned int t = temp[idx1];
temp[idx1] = temp[idx2];
temp[idx2] += t;
}
}

__syncthreads();
if(2*index + 1 < array_size) {
d_cdf[2*index] = temp[2*index];
d_cdf[2*index + 1] = temp[2*index + 1];
}
}