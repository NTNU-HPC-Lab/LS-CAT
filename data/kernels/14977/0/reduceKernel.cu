#include "hip/hip_runtime.h"
#include "includes.h"

// Number of elements to put in the test array
#define TEST_SIZE 16
#define NUM_BINS 10

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 4
// Functions to reduce with
#define ADD 0
#define MIN 1
#define MAX 2
// Device functions

__device__ float maxOp(float a, float b) {
return a > b ? a : b;
}
__device__ float minOp(float a, float b) {
return a < b ? a : b;
}
__device__ float addOp(float a, float b) {
return a + b;
}
__global__ void reduceKernel(float* array, const size_t array_size, const unsigned int op, const size_t step)
{
__shared__ float temp[BLOCK_WIDTH];
int bx = blockIdx.x;
int tx = threadIdx.x;
int index = BLOCK_WIDTH * bx + tx;

if(index < array_size) {
temp[tx] = array[index * step];
}

__syncthreads();

// Reduce
for(int offset = BLOCK_WIDTH >> 1; offset > 0; offset >>= 1) {
if(tx < offset) {
switch(op) {
case ADD:
temp[tx] = addOp(temp[tx], temp[tx + offset]);
break;
case MIN:
temp[tx] = minOp(temp[tx], temp[tx + offset]);
break;
case MAX:
temp[tx] = maxOp(temp[tx], temp[tx + offset]);
break;
default:
break;
}
}
__syncthreads();
}

if(index < array_size) {
array[BLOCK_WIDTH * bx] = temp[0];
}

}