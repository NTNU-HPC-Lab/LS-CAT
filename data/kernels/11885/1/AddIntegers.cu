#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddIntegers(int *arr1, int *arr2, int num_elements)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < num_elements)
{
arr1[id] += arr2[id];
}
}