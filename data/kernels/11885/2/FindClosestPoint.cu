#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindClosestPoint(float3 *points, int *closestPoint, const int numberPoints)
{
// used to identify the thread that is currently running
int idx = blockIdx.x * blockDim.x + threadIdx.x;
// now find the closest point to each point
// 'i' represents the current point that we are finding the closest point to!
int distanceBetweenPoints = 9999999, tempDistance = 0;
for (int j = 0; j < numberPoints; j++)
if (idx != j)		// dont check the distance between the point and itself
{
tempDistance = pow((points[idx].x - points[j].x), 2) + pow((points[idx].y - points[j].y), 2);
if (tempDistance < distanceBetweenPoints)
{
distanceBetweenPoints = tempDistance;
closestPoint[idx] = j;
}
}
}