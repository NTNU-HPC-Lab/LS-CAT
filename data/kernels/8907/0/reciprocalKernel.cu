#include "hip/hip_runtime.h"
#include "includes.h"
/*
============================================================================
Name        : Teste.cu
Author      :
Version     :
Copyright   : Your copyright notice
Description : CUDA compute reciprocals
============================================================================
*/


static void CheckCudaErrorAux(const char *, unsigned, const char *,
hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* CUDA kernel that computes reciprocal values for a given vector
*/

/**
* Host function that copies the data and launches the work on GPU
*/
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < vectorSize)
data[idx] = 1.0 / data[idx];
}