#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( int *a, int dimx, int dimy ) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
a[i] = blockIdx.x * dimx + threadIdx.x;
}