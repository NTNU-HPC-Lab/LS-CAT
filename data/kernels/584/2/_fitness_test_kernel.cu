#include "hip/hip_runtime.h"
#include "includes.h"





__device__ int fitness_fun(unsigned char* actual_genes, unsigned char* expected_genes, int genes_len) {
if (actual_genes == NULL) {
printf("actual genes are NULL\n");
return 1;
}
if (expected_genes == NULL) {
printf("expected_genes are NULL\n");
return 2;
}
//unsigned char* expected_genes; // TODO: this should be the target BMP image.
int match_counter = 0;
for (int geneIndex = 0; geneIndex < genes_len; geneIndex++) {
unsigned char a_byte = expected_genes[geneIndex];
unsigned char b_byte = actual_genes[geneIndex];
unsigned char res_byte = ~(a_byte ^ b_byte); // ^ is xor, 1 if both bits are different. That negated, are the common bits

// now count bits in 1
unsigned int res_int = (unsigned int) res_byte;
match_counter += __popc(res_int); // count bits in 1 in cuda

}
return match_counter;
}
__global__ void _fitness_test_kernel(unsigned char* expected_genes, unsigned char* actual_genes, const int genes_size, int* ftnss_ptr) {
if (blockIdx.x == 0) {
if (threadIdx.x == 0) {
*ftnss_ptr = fitness_fun(actual_genes, expected_genes, genes_size);
}
}
}