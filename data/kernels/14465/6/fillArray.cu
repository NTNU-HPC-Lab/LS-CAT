#include "hip/hip_runtime.h"
#include "includes.h"



#define A 1.2f
#define B 0.5f
#define MIN_LEARNING_RATE 0.000001f
#define MAX_LEARNING_RATE 50.0f

// Device functions

// Array[height * width]
__global__ void fillArray(float *array, float value, int arrayLength)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= arrayLength)
return;

array[i] = value;
}