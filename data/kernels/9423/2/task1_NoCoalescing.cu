#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void task1_NoCoalescing(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
auto index = blockIdx.x * blockDim.x + threadIdx.x + 7;
if (index > size + 6) {
return;
}
if (index >= size) {
index -= 7;
}
result[index] = a[index] * b[index];
}