#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyGlobal(unsigned const* left, unsigned const* right, unsigned* result, size_t size)
{
auto row = blockIdx.y * blockDim.y + threadIdx.y;
auto col = blockIdx.x * blockDim.x + threadIdx.x;
if (row < size && col < size) {
auto sum = 0u;
for (int k = 0; k < size; k++) {
sum += left[row * size + k] * right[k * size + col];
}
result[row * size + col] = sum;
}
}