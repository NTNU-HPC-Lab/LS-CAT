#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void task1_Coalescing(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
auto index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= size) {
return;
}
result[index] = a[index] * b[index];
}