#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void task2Kernel(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
auto index = blockIdx.x * blockDim.x + threadIdx.x;
if (index >= size) {
return;
}
result[index] = a[index] * b[index];
}