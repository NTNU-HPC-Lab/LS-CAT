#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexAdd( float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim )
{
int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

long flat_size = src_size / idx_size;

if (thread_idx < flat_size)
{
long coeff = 0;
for (int i=0; i<idx_size; i++)
{
int leftover = thread_idx;
int targetIdx = 0;
int resIdx = 0;
for (int d=0; d<res_nDim; d++)
{
if (d < dim)
{
long stride_d = res_stride[d] / size_dim;
coeff = leftover / stride_d;
leftover -= coeff * stride_d;
targetIdx += coeff * stride_d * idx_size;
resIdx += coeff * res_stride[d];
}
else if (d > dim)
{
coeff = leftover / res_stride[d];
leftover -= coeff * res_stride[d];
targetIdx += coeff * res_stride[d];
resIdx += coeff * res_stride[d];
}
}
atomicAdd(&res[resIdx + ((long)(index[i])-1)*res_stride[dim]], src[targetIdx + i*res_stride[dim]]);
}
}
}