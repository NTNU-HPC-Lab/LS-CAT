#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float* A, const float* B, float* C, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < N)
C[i] = A[i] + B[i];
}