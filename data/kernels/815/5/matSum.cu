#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matSum(float* S, float* A, float* B, int N) {
int j = blockIdx.y*blockDim.y + threadIdx.y;
int i = blockIdx.x*blockDim.x + threadIdx.x;
int tid = i*N + j;
if (tid < N*N) {
S[tid] = A[tid] + B[tid];
}
}