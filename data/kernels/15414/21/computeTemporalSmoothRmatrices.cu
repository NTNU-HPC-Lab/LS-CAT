#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;

if (sampleIdx < numSamples)
{
int subArraySizeLocal = subArraySizes[sampleIdx];
int numelR = subArraySizeLocal*subArraySizeLocal;
int numelRfull = subArraySize*subArraySize;

int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing));
int lastIdx = min((int)(numSamples)-1, sampleIdx + (int)(temporalSmoothing));

float scaling = 1.0f;
for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x*blockDim.y)
{
int colIdx = matrixIdx % subArraySizeLocal;
int rowIdx = matrixIdx / subArraySizeLocal;
int matrixStorageIdx = colIdx + rowIdx * subArraySize;

float finalEntry = 0.0f;
for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
{
finalEntry += Rmatrices[matrixStorageIdx + tempIdx*numelRfull];
}
TempRmatrices[matrixStorageIdx + sampleIdx*numelRfull] = finalEntry*scaling;
}
}
}