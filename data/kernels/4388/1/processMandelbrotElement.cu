#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int doIterations(double const realPart0, double const imagPart0, unsigned int const maxIters) {
// Initialise: z = z0
double realPart = realPart0;
double imagPart = imagPart0;
unsigned int count = 0;
// Loop until escape
while ((count <= maxIters)
&& ((realPart*realPart + imagPart * imagPart) <= 4.0)) {
++count;
// Update: z = z*z + z0;
double const oldRealPart = realPart;
realPart = realPart * realPart - imagPart * imagPart + realPart0;
imagPart = 2.0*oldRealPart*imagPart + imagPart0;
}
return count;
}
__device__ size_t calculateGlobalIndex() {
// Which block are we?
size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
// Which thread are we within the block?
size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
// How big is each block?
size_t const threadsPerBlock = blockDim.x*blockDim.y;
// Which thread are we overall?
return localThreadIdx + globalBlockIndex * threadsPerBlock;

}
__global__ void processMandelbrotElement( double * out, const double * x, const double * y, const unsigned int maxIters, const unsigned int numel) {
// Work out which thread we are
size_t const globalThreadIdx = calculateGlobalIndex();

// If we're off the end, return now
if (globalThreadIdx >= numel) {
return;
}

// Get our X and Y coords
double const realPart0 = x[globalThreadIdx];
double const imagPart0 = y[globalThreadIdx];

// Run the itearations on this location
unsigned int const count = doIterations(realPart0, imagPart0, maxIters);
out[globalThreadIdx] = log(double(count + 1));
}