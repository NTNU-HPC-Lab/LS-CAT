#include "hip/hip_runtime.h"
#include "includes.h"



#define Columnas 10
#define Filas 10
hipError_t addWithCuda(int* c, const int* a, unsigned int size);


__device__ unsigned int computeOutputEdge(int mask[][3], int vecinos[][3], int rows, int cols) {

float result = 1;
int sum = 0;

for (int i = 0; i < rows; i++) {
for (int j = 0; j < cols; j++) {
float mul = mask[i][j] * vecinos[i][j];
sum = sum + mul;
}
}
result = abs(sum);
return (int)result;
}
__global__ void bordes(int* val2, int* val1, int m, int n)
{

int column = threadIdx.x + blockDim.x * blockIdx.x;
int row = threadIdx.y + blockDim.y * blockIdx.y;

int myEdge[3][3] = { {0,1,0},{1,-4,1},{0,1,0} };
//int filas = (sizeof(myMask)/sizeof(myMask[0]));

if (row < m && column < n) {

int thread_id1 = (row - 1) * n + (column - 1);
int thread_id2 = (row - 1) * n + (column);
int thread_id3 = (row - 1) * n + (column + 1);

int thread_id4 = (row)* n + (column - 1);

int thread_id5 = (row)* n + (column);

int thread_id6 = (row)* n + (column + 1);

int thread_id7 = (row + 1) * n + (column - 1);
int thread_id8 = (row + 1) * n + (column);
int thread_id9 = (row + 1) * n + (column + 1);

//int my_val = val1[thread_id5];

//printf("row: %d, \tcol: %d, \tvalor: %d\n", row, column, my_val);

val2[thread_id5] = val1[thread_id5];

if ((row > 0 && row < (m - 1)) && (column > 0 && column < (n - 1)))
{
int my_val0 = val1[thread_id1];
int my_val2 = val1[thread_id2];
int my_val3 = val1[thread_id3];
int my_val4 = val1[thread_id4];
int my_val5 = val1[thread_id5]; //doubly-subscripted access
int my_val6 = val1[thread_id6];
int my_val7 = val1[thread_id7];
int my_val8 = val1[thread_id8];
int my_val9 = val1[thread_id9];
//printf("row: %d, col: %d, value: %d\n", row, column, my_val);

int myMask2[3][3] = { {(my_val0),(my_val2),(my_val3)},
{(my_val4),(my_val5),(my_val6)},
{(my_val7),(my_val8),(my_val9)} };

unsigned int output = computeOutputEdge(myEdge, myMask2, 3, 3);
//printf("row: %d,\t col: %d,\t Valor Original: %d,\t Nuevo Valor: %d\n", row, column, my_val5,output);
//printf("Salida: %d \n", output);
//printf("Entro\n");
val2[thread_id5] = output;
}
}
}