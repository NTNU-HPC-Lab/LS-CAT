#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {

//@@ Using privitization technique
__shared__ unsigned int hist[NUM_BINS];

int numOfElementsPerThread = NUM_BINS / BLOCK_SIZE;

int i = blockDim.x * blockIdx.x + threadIdx.x;

for (int j = 0; j < numOfElementsPerThread; ++j)
hist[threadIdx.x + blockDim.x*j] = 0;

__syncthreads();

if (i < num_elements)
atomicAdd(&hist[input[i]], 1);
__syncthreads();

for (int k = 0; k < numOfElementsPerThread; ++k)
atomicAdd(&bins[threadIdx.x + blockDim.x*k], hist[threadIdx.x+blockDim.x*k]);
}