#include "hip/hip_runtime.h"
#include "includes.h"


#define TILE_WIDTH = 16;
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {

//@@Y-axis matrix dimension
int row = blockIdx.y*blockDim.y + threadIdx.y;

//@@X-axis matrix Dimension
int columns = blockIdx.x*blockDim.x + threadIdx.x;

//@@Initilaizing final value to add in the output matrix
float pValue = 0;

//@@checking for boundary condition
if (row < numARows  && columns < numBColumns) {

//@@adding values from 0 to matrix A width and from 0 to martrix B Height
for (int k = 0; k < numAColumns; k++) {

//@@Summation of the rows from matrix A and columns from matrix B to pValue
pValue += A[row*numAColumns + k] * B[k*numBColumns + columns];

}

//@@add the final value to the output matrix
C[row*numCColumns + columns] = pValue;

}

}