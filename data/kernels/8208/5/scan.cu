#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float *input, float *output, float *aux, int len) {

//@@declaring shared memeory of size 2*inputSize
__shared__ float XY[2 * BLOCK_SIZE];

//@@X-axis block id
int bx = blockIdx.x;

//@@X-axis thread id
int tx = threadIdx.x;

int i = 2 * bx * blockDim.x + tx;

//@@ loading data from global memory to shared memory stage 1
if (i<len)
XY[tx] = input[i];

//@@ loading data from global memory to shared memory stage 2
if (i + blockDim.x<len)
XY[tx + blockDim.x] = input[i + blockDim.x];

//@@making sure that all threads in a block are done with loading data from global memory to shared memory
//@@before proceeding to the calculations phase
__syncthreads();

for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2){
//@@making sure that all threads in a block are done with previous step before starting the next
__syncthreads();

int index = (tx + 1)*stride * 2 - 1;

if (index < 2 * BLOCK_SIZE)
XY[index] += XY[index - stride];
}

for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
//@@making sure that all threads in a block are done with previous step before starting the next
__syncthreads();

int index = (tx + 1)*stride * 2 - 1;

if (index + stride < 2 * BLOCK_SIZE)
XY[index + stride] += XY[index];
}

//@@making sure that all threads in a block are done with previous step before starting the next
__syncthreads();

if (i < len)
output[i] = XY[tx];

if (i + blockDim.x < len)
output[i + blockDim.x] = XY[tx + blockDim.x];

//@@storing the block sum to the aux array
if (aux != NULL && tx == 0)
aux[bx] = XY[2 * blockDim.x - 1];
}