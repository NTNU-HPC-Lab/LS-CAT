#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addScannedBlockSums(float *input, float *aux, int len) {

int tx = threadIdx.x;

int bx = blockIdx.x;

int dx = blockDim.x;

int i = 2 * bx * dx + tx;

if (bx > 0) {

if (i < len)
aux[i] += input[bx-1];

if (i + dx < len)
aux[i + dx] += input[blockIdx.x - 1];
}
}