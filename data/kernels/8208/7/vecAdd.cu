#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float *in1, float *in2, float *out, int len) {

int i = threadIdx.x + (blockDim.x * blockIdx.x);

//@@checking boundary condition and adding vectors
if (i < len)
out[i] = in1[i] + in2[i];
}