#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saturate(unsigned int *bins, unsigned int num_bins) {

//@@If the bin value is more than 127, make it equal to 127
for (int i = 0; i < NUM_BINS / BLOCK_SIZE; ++i)

if (bins[threadIdx.x + blockDim.x*i] >= 128)

bins[threadIdx.x + blockDim.x*i]  = 127;
}