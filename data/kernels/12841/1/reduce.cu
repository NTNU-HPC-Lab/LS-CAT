#include "hip/hip_runtime.h"
#include "includes.h"

//Macros
#define min(a, b) ( (a)<(b)? (a): (b) )
#define max(a, b) ( (a)>(b)? (a): (b) )

//Constants
#define MAX_VECTOR_COUNT 5

//Vector structure
typedef struct {
float e[3];
}Vec3f;

//Global array
Vec3f vecArray[MAX_VECTOR_COUNT];
Vec3f newvecArray[MAX_VECTOR_COUNT];

//forward declarations

__global__ void reduce(Vec3f *input, Vec3f *output){
extern __shared__ Vec3f sdata[];

// each thread loadsome element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i   = threadIdx.x + blockIdx.x * blockDim.x;
sdata[tid] = input[i];
__syncthreads();

//perform reduction in shared mem
for(unsigned int s=1; s < blockDim.x; s *= 2) {
//int s = 2;
if(tid % (2*s) == 0){

sdata[tid].e[0] += sdata[tid + s].e[0];	//summing
sdata[tid].e[1] += sdata[tid + s].e[1];
sdata[tid].e[2] += sdata[tid + s].e[2];
/*
sdata[tid].e[0] = min( sdata[tid].e[0], sdata[tid + s].e[0] );	//min
sdata[tid].e[1] = min( sdata[tid].e[1], sdata[tid + s].e[1] );
sdata[tid].e[2] = min( sdata[tid].e[2], sdata[tid + s].e[2] );

sdata[tid].e[0] = max( sdata[tid].e[0], sdata[tid + s].e[0] );	//max
sdata[tid].e[1] = max( sdata[tid].e[1], sdata[tid + s].e[1] );
sdata[tid].e[2] = max( sdata[tid].e[2], sdata[tid + s].e[2] );
*/
}
__syncthreads();
}

// write result for this block to global mem
if(tid == 0) output[blockIdx.x] = sdata[0];
}