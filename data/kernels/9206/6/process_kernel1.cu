#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process_kernel1(const float* input1, const float* input2, float* output, int numElements){

int blockNum = blockIdx.z*(gridDim.x*gridDim.y) + blockIdx.y*gridDim.x + blockIdx.x;
int threadNum = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int globalThreadId = blockNum*(blockDim.x*blockDim.y*blockDim.z) + threadNum;

if(globalThreadId < numElements)
output[globalThreadId] = (float)sin(input1[globalThreadId]) + (float)cos(input2[globalThreadId]);

}