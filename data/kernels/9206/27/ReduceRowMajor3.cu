#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceRowMajor3(int *g_idata, int *g_odata, int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
sdata[tid] = 0;
if(i < size)
sdata[tid] = g_idata[i];
__syncthreads();
for(unsigned int s = blockDim.x/2; s > 3; s/=2) {
if(tid < s) {
sdata[tid] += sdata[tid+s];
}
__syncthreads();
}
if(tid == 0) {
g_odata[blockIdx.x*4] = sdata[0];
g_odata[blockIdx.x*4+1] = sdata[1];
g_odata[blockIdx.x*4+2] = sdata[2];
g_odata[blockIdx.x*4+3] = sdata[3];
}
}