#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution1D(const int *d_arr, const int *d_conv, int *d_result, int N, int M) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int globalId = i*N + j;
if(globalId < N) {
int convSum = 0, cnum = 0, k = M/2;
for(int i=-k; i<=k; i++) {
if(globalId + i >= 0 && globalId + i < N && cnum < M) {
convSum += d_arr[globalId + i]*d_conv[cnum];
}
cnum++;
}
d_result[globalId] = convSum;
}
}