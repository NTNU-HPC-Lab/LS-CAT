#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void warpReduce(volatile int* sdata, int tid, int n) {
if(tid + 32 < n)
sdata[tid] += sdata[tid+32];
if(tid + 16 < n)
sdata[tid] += sdata[tid+16];
if(tid + 8 < n)
sdata[tid] += sdata[tid+8];
if(tid + 4 < n)
sdata[tid] += sdata[tid+4];
}
__global__ void ReduceRowMajor5(int *g_idata, int *g_odata, int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
sdata[tid] = 0;
if(i < size)
sdata[tid] = g_idata[i];
__syncthreads();
for(unsigned int s = blockDim.x/2; s >= 32; s/=2) {
if(tid < s) {
sdata[tid] += sdata[tid+s];
}
__syncthreads();
}
if(tid < 32) {
warpReduce(sdata, tid, size);
}
if(tid == 0) {
g_odata[blockIdx.x*4] = sdata[0];
g_odata[blockIdx.x*4+1] = sdata[1];
g_odata[blockIdx.x*4+2] = sdata[2];
g_odata[blockIdx.x*4+3] = sdata[3];
}
}