#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution2D(const float *d_arr, const float *d_mask, float *d_result, int N) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int globalId = i*N + j;
if(i < N && j< N) {
float avgSum = 0;
int id, cnum = 0;
for(int p = i-1; p <= i+1; p++) {
for(int q = j-1; q<= j+1; q++) {
if(p >=0 && p < N && q>=0 && q < N) {
id = p*N + q;
avgSum += d_arr[id]*d_mask[cnum];
}
cnum++;
}
}
d_result[globalId] = avgSum;
}
}