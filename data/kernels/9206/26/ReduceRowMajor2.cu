#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceRowMajor2(int *g_idata, int *g_odata, int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
sdata[tid] = 0;
if(i < size)
sdata[tid] = g_idata[i];
__syncthreads();
for(unsigned int s = 4; s < blockDim.x; s*=2) {
int index = 2*s*tid;
if(index < blockDim.x) {
sdata[index] += sdata[index+s];
sdata[index+1] += sdata[index+s+1];
sdata[index+2] += sdata[index+s+2];
sdata[index+3] += sdata[index+s+3];
}
__syncthreads();
}
if(tid == 0) {
g_odata[blockIdx.x*4] = sdata[0];
g_odata[blockIdx.x*4+1] = sdata[1];
g_odata[blockIdx.x*4+2] = sdata[2];
g_odata[blockIdx.x*4+3] = sdata[3];
}
}