#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceRowMajor(int *g_idata, int *g_odata, int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
sdata[tid] = 0;
if(i < size)
sdata[tid] = g_idata[i];
__syncthreads();
for(unsigned int s = 4; s < blockDim.x; s*=2) {
if(tid%(2*s) == 0) {
sdata[tid] += sdata[tid+s];
sdata[tid+1] += sdata[tid+s+1];
sdata[tid+2] += sdata[tid+s+2];
sdata[tid+3] += sdata[tid+s+3];
}
__syncthreads();
}
if(tid == 0) {
g_odata[blockIdx.x*4] = sdata[0];
g_odata[blockIdx.x*4+1] = sdata[1];
g_odata[blockIdx.x*4+2] = sdata[2];
g_odata[blockIdx.x*4+3] = sdata[3];
}
}