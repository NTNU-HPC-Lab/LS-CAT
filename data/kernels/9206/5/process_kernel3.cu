#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process_kernel3(const float* input, float* output, int numElements){

int blockNum = blockIdx.z*(gridDim.x*gridDim.y) + blockIdx.y*gridDim.x + blockIdx.x;
int threadNum = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int globalThreadId = blockNum*(blockDim.x*blockDim.y*blockDim.z) + threadNum;

if(globalThreadId < numElements)
output[globalThreadId] = (float)sqrt(input[globalThreadId]);
}