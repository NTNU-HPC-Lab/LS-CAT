#include "hip/hip_runtime.h"
#include "includes.h"

// Device Function for process_kernel1


// Device Function for process_kernel2


// Device Function for process_kernel3

__global__ void process_kernel3(const float* input, float* output, int numElements){

int blockNum = blockIdx.z*(gridDim.x*gridDim.y) + blockIdx.y*gridDim.x + blockIdx.x;
int threadNum = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int globalThreadId = blockNum*(blockDim.x*blockDim.y*blockDim.z) + threadNum;

if(globalThreadId < numElements)
output[globalThreadId] = (float)sqrt(input[globalThreadId]);
}