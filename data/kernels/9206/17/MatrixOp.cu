#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixOp(int *arr, int N) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int swapVar;
if(i<N && j<N) {
if(j%2==0 && (j+1)!=N) {
// swap elements
swapVar = arr[i*N + j];
arr[i*N + j] = arr[i*N+j+1];
arr[i*N+j+1] = swapVar;
}
__syncthreads();
if(i > j){
arr[j*N + i] = arr[i*N+j];
}
}
}