#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arradd(const int *md, const int *nd, int *pd, int size){
int myid = blockDim.x*blockIdx.x + threadIdx.x;
if(myid < size)
pd[myid] = md[myid] + nd[myid];
}