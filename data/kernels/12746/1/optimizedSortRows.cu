#include "hip/hip_runtime.h"
#include "includes.h"

#define THRESHOLD 10010000

__device__ int cudaGetNextInColor(int *image, int x, int row, int imageWidth, int color){
for (int i = x + 1; i < imageWidth; ++i)
{
if(THRESHOLD >= (color - image[row*imageWidth + i])){
return i-1;
}
}
return imageWidth - 1;
}
__device__ int cudaGetFirstNotInColor(int *image, int x, int row, int imageWidth, int color){
for (int i = x; i < imageWidth; ++i)
{
if(THRESHOLD < (color - image[row*imageWidth + i])){
return i;
}
}
return -1;
}
__device__ void optimizedBubbleSort(int *pixelsToSort, int length){
for(int i = 0; i < length; i++ )
{
for(int j = 0; j < length-1; j++)
{
if( pixelsToSort[j] > pixelsToSort[j+1]){
pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
pixelsToSort[j+1] = pixelsToSort[j] ^ pixelsToSort[j+1];
pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
}
}
}
}
__global__ void optimizedSortRows(int *image, int imageHeight, int imageWidth, int colorMode){
int row = blockIdx.x * blockDim.x + threadIdx.x;
if(row < imageHeight)
{
int startingX = 0;
int finishX = 0;
int *pixelsToSort = new int[1024];

while(finishX < imageWidth)
{
startingX = cudaGetFirstNotInColor(image, startingX, row, imageWidth, colorMode);
finishX = cudaGetNextInColor(image, startingX, row, imageWidth, colorMode);

if(startingX < 0)
break;

int pixelsToSortLength = (finishX - startingX < 1024) ? finishX - startingX : 1024;

for (int i = 0; i < pixelsToSortLength; ++i)
{
pixelsToSort[i] = image[row*imageWidth + startingX + i];
}

optimizedBubbleSort(pixelsToSort, pixelsToSortLength);

for (int i = 0; i < pixelsToSortLength; ++i)
{
image[row*imageWidth + startingX + i] = pixelsToSort[i];
}

startingX = finishX + 1;
}

free(pixelsToSort);
}
}