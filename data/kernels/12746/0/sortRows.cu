#include "hip/hip_runtime.h"
#include "includes.h"

#define THRESHOLD 10010000

__device__ int cudaGetNextInColor(int *image, int x, int row, int imageWidth, int color){
for (int i = x + 1; i < imageWidth; ++i)
{
if(THRESHOLD >= (color - image[row*imageWidth + i])){
return i-1;
}
}
return imageWidth - 1;
}
__device__ int cudaGetFirstNotInColor(int *image, int x, int row, int imageWidth, int color){
for (int i = x; i < imageWidth; ++i)
{
if(THRESHOLD < (color - image[row*imageWidth + i])){
return i;
}
}
return -1;
}
__device__ void bubbleSort(int *pixelsToSort, int length){
for(int i = 0; i < length; i++ )
{
for(int j = 0; j < length-1; j++)
{
if( pixelsToSort[j] > pixelsToSort[j+1]){
int tmp = pixelsToSort[j];
pixelsToSort[j] = pixelsToSort[j+1];
pixelsToSort[j+1] = tmp;
}
}
}
}
__global__ void sortRows(int *image, int imageHeight, int imageWidth, int colorMode){
int row = blockIdx.x * blockDim.x + threadIdx.x;
if(row < imageHeight)
{
int startingX = 0;
int finishX = 0;

while(finishX < imageWidth)
{
startingX = cudaGetFirstNotInColor(image, startingX, row, imageWidth, colorMode);
finishX = cudaGetNextInColor(image, startingX, row, imageWidth, colorMode);

if(startingX < 0)
break;

bubbleSort(&image[row*imageWidth + startingX], finishX - startingX);

startingX = finishX + 1;
}
}
}