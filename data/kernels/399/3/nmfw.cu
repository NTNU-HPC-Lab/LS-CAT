#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nmfw(double *a, int r, int c, int k, double *w, double *h, double *wcp)//must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

//compute W
if (col < k && row < r) {
//ah'
double sum = 0.0;
double temp = 0.0;
for (int i = 0; i < c; i++)
sum += a[row*c + i]*h[col*c + i];
temp = w[row*k+col]*sum;
//whh'
sum = 0.0;
for (int i = 0; i < c; i++) {
for (int j = 0; j < k; j++) {
sum += w[row*k + j]*h[j*c + i]*h[col*c+i];
}
}
__syncthreads();
wcp[row*k+col] = temp/sum;
}
}