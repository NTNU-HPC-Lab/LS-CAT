#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nmfh(double *a, int r, int c, int k, double *w, double *h, double *hcp)//must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

//compute H
if (row < k && col < c) {
//w'a
double temp = 0.0;
double sum;
sum = 0.0;
for (int i = 0; i < r; i++)
sum += w[i*k + row]*a[i*c+col];

temp = h[row*c+col]*sum;
//w'wh
sum = 0.0;
for (int i = 0; i < k; i++)
for (int j = 0; j < r; j++)
sum += w[j*k + row]*w[j*k + i]*h[i*c+col];

__syncthreads();
hcp[row*c+col] = temp/sum;
}
}