#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

//test code
__global__ void nmfw(float *a, int r, int c, int k, float *w, float *h, float *wcp)//must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

//compute W
if (col < k && row < r) {
//ah'
float sum = 0.0;
float temp = 0.0;
for (int i = 0; i < c; i++)
sum += a[row*c + i]*h[col*c + i];
temp =  w[row*k+col]*sum;
//whh'
sum = 0.0;
for (int i = 0; i < c; i++) {
for (int j = 0; j < k; j++) {
sum += w[row*k + j]*h[j*c + i]*h[col*c+i];
}
}
__syncthreads();
wcp[row*k+col] = temp/sum;
}
}