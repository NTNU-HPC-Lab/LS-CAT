#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void quad(float *a, int n, float *u, float *v)
{
int col  = blockIdx.x * blockDim.x + threadIdx.x;
int row  = blockIdx.y * blockDim.y + threadIdx.y;

if (row < n && col < n && col >= row) {
float sum = u[col]*a[row*n+col]*u[row];
if (col == row)
atomicAdd(v, sum);
else
atomicAdd(v, 2*sum);
}
}