#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nmfcpy(double *mat, double *matcp, int m, int n) //kernel copy must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

if (row < m && col < n)
mat[row*n+col] = matcp[row*n+col];
}