#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

//test code
__global__ void nmfcpy(float *mat, float *matcp, int m, int n) //kernel copy must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

if (row < m && col < n)
mat[row*n+col] = matcp[row*n+col];
}