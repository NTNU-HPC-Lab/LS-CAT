#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

//test code
__global__ void nmfh(float *a, int r, int c, int k, float *w, float *h, float *hcp)//must be block synchronized!!!
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

//compute H
if (row < k && col < c) {
//w'a
float temp = 0.0;
float sum;
sum = 0.0;
for (int i = 0; i < r; i++)
sum += w[i*k + row]*a[i*c+col];

temp = h[row*c+col]*sum;
//w'wh
sum = 0.0;
for (int i = 0; i < k; i++)
for (int j = 0; j < r; j++)
sum += w[j*k + row]*w[j*k + i]*h[i*c+col];

__syncthreads();
hcp[row*c+col] = temp/sum;
}
}