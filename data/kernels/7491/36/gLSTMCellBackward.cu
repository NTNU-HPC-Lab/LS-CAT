#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gLSTMCellBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float m = !mask || mask[j];

float* rowOutCell = outCell + j * cols;
float* rowOutXW = outXW + j * cols * 4;
float* rowOutSU = outSU + j * cols * 4;

const float* rowCell = cell + j * cols;
const float* xWrow = xW + j * cols * 4;
const float* sUrow = sU + j * cols * 4;

const float* rowAdj = adj + j * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
float gf = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);

int k = i + cols;
float gi = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

int l = i + 2 * cols;
float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

float adj = rowAdj[i];

// dc/dc_{t-1}
if(outCell)
rowOutCell[i] += (m * gf - m + 1) * adj;

// dc/d(b_f) = dc/d(xW_f) ...
float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
if(outXW)
rowOutXW[i] += dcdxf;
if(outSU)
rowOutSU[i] += dcdxf;
if(outB)
atomicAdd(outB + i, dcdxf);

// dc/d(b_i) ...
float dcdb_i = m * gc * gi * (1 - gi) * adj;
if(outXW)
rowOutXW[k] += dcdb_i;
if(outSU)
rowOutSU[k] += dcdb_i;
if(outB)
atomicAdd(outB + k, dcdb_i);

// dc/d(b_c) ...
float dcdxc = m * gi * (1 - gc * gc) * adj;
if(outXW)
rowOutXW[l] += dcdxc;
if(outSU)
rowOutSU[l] += dcdxc;
if(outB)
atomicAdd(outB + l, dcdxc);
}
}
}
}
}