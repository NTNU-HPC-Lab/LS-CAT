#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gPasteCols(float* out, const float* in, size_t rows, size_t colsOut, const size_t* targetColIdx, size_t colsIn) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
const float* rowIn = in + j * colsIn;
float* rowOut = out + j * colsOut;

for(int tid = 0; tid < colsIn; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < colsIn)
rowOut[targetColIdx[i]] += rowIn[i];
}
}
}
}