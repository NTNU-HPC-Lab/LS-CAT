#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gAddRow(float* out, const float* in, int length) {
for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
if(index < length) {
out[index] = in[index] + out[index];
}
}
}