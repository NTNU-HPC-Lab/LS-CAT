#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void vectorAdd(const int *a, const int *b, int *c, int N)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

while(tid < N)
{
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}
}