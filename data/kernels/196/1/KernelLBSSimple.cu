#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelLBSSimple(int aCount, const int* b_global, int bCount, int* indices_global) {

__shared__ int data_shared[NT * VT];

int tid = threadIdx.x;

// Load bCount elements from B into data_shared.
int x[VT];
#pragma unroll
for(int i = 0; i < VT; ++i) {
int index = NT * i + tid;
if(index < bCount) x[i] = b_global[index];
}

#pragma unroll
for(int i = 0; i < VT; ++i)
data_shared[NT * i + tid] = x[i];
__syncthreads();

// Each thread searches for its Merge Path partition.
int diag = VT * tid;
int begin = max(0, diag - bCount);
int end = min(diag, aCount);

while(begin < end) {
int mid = (begin + end)>> 1;
int aKey = mid;
int bKey = data_shared[diag - 1 - mid];
bool pred = aKey < bKey;
if(pred) begin = mid + 1;
else end = mid;
}
int mp = begin;

// Sequentially search, comparing indices a to elements data_shared[b].
// Store indices for A in the right-side of the shared memory array.
// This lets us complete the search in just a single pass, rather than
// the search and compact passes of the generalized vectorized sorted
// search function.
int a = mp;
int b = diag - a;

#pragma unroll
for(int i = 0; i < VT; ++i) {
bool p;
if(b >= bCount) p = true;
else if(a >= aCount) p = false;
else p = a < data_shared[b];

if(p)
// If a < data_shared[b], advance A and store the index b - 1.
data_shared[bCount + a++] = b - 1;
else
// Just advance b.
++b;
}
__syncthreads();

// Store all indices to global memory.
for(int i = tid; i < aCount; i += NT)
indices_global[i] = data_shared[bCount + i];
}