#include "hip/hip_runtime.h"
#include "includes.h"
#define N 100


__global__ void add(int *a, int *c)
{
int tID   = blockIdx.x;
if (tID < N)
{
c[tID] = 3*a[tID];
}
}