#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elevar_al_cuadrado(float * d_salida, float * d_entrada){
int idx = threadIdx.x;
float f = d_entrada[idx];
d_salida[idx] = f*f;
}