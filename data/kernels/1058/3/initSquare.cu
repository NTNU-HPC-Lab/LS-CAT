#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initSquare(float* a, float* x, float totalX, int n, int ghosts){
int i = threadIdx.x + blockDim.x*blockIdx.x;
for(int j = 0; blockDim.x*j + i < n; j++){
int index = j*blockDim.x+i;
if(index > n/3 && index < 2*n/3)
a[index+ghosts] = 1.5;
else a[index+ghosts] = .5;
}
__syncthreads();

if(i==0){	//copy over for boundary conditions
for(int j = 0; j < ghosts; j++){
a[j] = a[j+n];
a[n+ghosts+j] = a[ghosts+j];
// a[j] = a[ghosts];
// a[n+ghosts+j] = a[n+ghosts-1];
}
// for(int z = 0; z < n+2*ghosts; z++){
// 	printf("%5d %10f\n", z, a[z]);
// }
}
}