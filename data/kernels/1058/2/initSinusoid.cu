#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initSinusoid(float* a, float* x, float totalX, int n, int ghosts, float shift, float amp){
int i = threadIdx.x + blockDim.x*blockIdx.x;
for(int j = 0; blockDim.x*j + i < n; j++){
int index = j*blockDim.x+i;
float temp = 0;
for(int z = 0; z < index; z++){
temp += x[z+ghosts];
}
a[index+ghosts] = sinpi((temp/totalX)*2)*amp + shift;
}
__syncthreads();

if(i==0){	//copy over for boundary conditions
for(int j = 0; j < ghosts; j++){
a[j] = a[j+n];
a[n+ghosts+j] = a[ghosts+j];
// a[j] = a[ghosts];
// a[n+ghosts+j] = a[n+ghosts-1];
}
// for(int z = 0; z < n+2*ghosts; z++){
// 	printf("%5d %10f\n", z, a[z]);
// }
}
}