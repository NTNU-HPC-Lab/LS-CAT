#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float colellaEvenInterp(float*in){
return (7.0/12)*(*(in+1) - *(in-1)) - (1.0/12)*((*(in+2) + *(in-1))-(*(in+1) + *(in-2)));
}
__device__ float linInterp(float* in){	//dangerous function, need to make sure you're only using it on the in-bounds parts of array
return ((*(in+1) + *in)/2) - ((*in + *(in-1))/2);
}
__global__ void advect(float* a, float* oA, float* x, float u, int n, int ghosts, float tmax){
__shared__ float dt;
__shared__ float minDx;
__shared__ float timeElapsed;
__shared__ int counter;
// __shared__ bool* areYouLessThan;

int i = threadIdx.x + blockDim.x*blockIdx.x;
timeElapsed = 0;

if(i == 0){
minDx = x[0];
dt = (minDx/u)/500;
counter = 0;
}
__syncthreads();

while(timeElapsed < tmax){
// if(i==0){
// 	minDx = x[0+ghosts];
// 	areYouLessThan = new bool[n];
// }
// __syncthreads();

// for(int j = 0; blockDim.x*j + i < n; j++){
// 	int index = j*blockDim.x+i;
// 	if(x[index+ghosts] < minDx)
// 		areYouLessThan[index] = true;
// 	else
// 		areYouLessThan[index] = false;
// }

// __syncthreads();

// if(i == 0){
// 	for(int j = 0; j < n; j++){
// 		if(areYouLessThan[j]){
// 			if(x[j+ghosts] < minDx)
// 				minDx = x[j+ghosts];
// 		}
// 	}

// dt = (minDx/u)/1000;
// 	// printf("dt: %f\n", dt);
// 	delete[] areYouLessThan;
// }

// __syncthreads();



for(int j = 0; blockDim.x*j + i < n; j++){
int index = j*blockDim.x+i+ghosts;
oA[index] = a[index] - dt*u*colellaEvenInterp(&a[index])/x[index];
// oA[index] = a[index] - dt*u*linInterp(&a[index])/x[index];
// printf("%d %f %f\n", index, a[index], oA[index]);
a[index] = oA[index];
}

__syncthreads();
// printf("%d here\n", i);
if(counter == 0){
for(int j = 0; blockDim.x*j + i < n; j++){
int index = j*blockDim.x+i+ghosts;
printf("%10f\t%10d\t%f\n", timeElapsed, index-ghosts, a[index]);
}
}
if(i==0){	//copy over for boundary conditions
for(int j = 0; j < ghosts; j++){
a[j] = a[j+n];
a[n+ghosts+j] = a[ghosts+j];
// a[j] = a[ghosts];
// a[n+ghosts+j] = a[n+ghosts-1];
}

// printf("%f\n",dt);
// printf("%p %f\n", dtt, *dtt);
timeElapsed += dt;
counter++;
if(counter == 10000)
counter = 0;
}
}
}