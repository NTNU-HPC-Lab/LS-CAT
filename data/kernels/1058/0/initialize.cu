#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize(float* a, float* oA, float* x, float totalSize, int n, int ghosts){
int i = threadIdx.x + blockDim.x*blockIdx.x;
for(int j = 0; blockDim.x*j + i < n + 2*ghosts; j++){
int index = blockDim.x*j + i;
a[index] = 0;
oA[index] = 0;
x[index] = totalSize/n;
}
}