#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void calculoAlgoritmoTroca(float *dev_matrizSuperior, int linhaPerm, int colunaPerm, int totalColunas, int totalLinhas)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
float fatorAnulador = 0.0;

//evitar operação em endereço invalido
//se for indice da linha permissivel, desconsiderar
if (i > totalLinhas || i == linhaPerm)
return;

//computar fator anulador da respectiva linha
fatorAnulador = dev_matrizSuperior[i * totalColunas + colunaPerm] * (-1);

//calcular os valores dos elementos da linha usando o fator anulador coletado
for (int coluna = 0; coluna < totalColunas; coluna++){

if (i * totalColunas + coluna > totalLinhas * totalColunas)
return;

//o valor da coluna permissivel sera 0
if (coluna == colunaPerm)
dev_matrizSuperior[i * totalColunas + coluna] = 0;
else
//os demais valores devem respeitar a equacao
//Valor = FatorAnulador * ValorRefLinhaPerm + LinhaAtual;
dev_matrizSuperior[i * totalColunas + coluna] = fatorAnulador
* dev_matrizSuperior[linhaPerm * totalColunas + coluna]
+ dev_matrizSuperior[i * totalColunas + coluna];
}

}