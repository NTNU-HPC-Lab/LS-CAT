#include "hip/hip_runtime.h"
#include "includes.h"
#define rows 1000
#define cols 1000

// CUDA kernel. Each thread takes care of one element of c

__global__ void matricesMul(double *m1, double *m2, double *m3)
{
// Get our global thread ID
int ti = blockIdx.y*blockDim.y+threadIdx.y;
int tj = blockIdx.x*blockDim.x+threadIdx.x;
// Make sure we do not go out of bounds
if(ti < rows && tj < cols){
double data= 0.0;
for(int k=0;k<rows;k++) data += m1[ti*rows+k] * m2[k*cols+tj];
m3[ti*rows+tj] = data;
}
}