#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(unsigned char *image,unsigned char *resImage,int rows,int cols){
/* it will modify each pixel */
//int ti = blockIdx.y*blockDim.y+threadIdx.y;
int tj = blockIdx.x*blockDim.x+threadIdx.x;
if(tj < rows*cols){
int pos = tj*chanDepth;
resImage[pos+BLUE] = image[pos+BLUE]*2;
resImage[pos+GREEN] = image[pos+GREEN]*2;
resImage[pos+RED] = image[pos+RED]*2;
}
}