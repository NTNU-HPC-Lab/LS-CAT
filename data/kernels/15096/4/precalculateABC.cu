#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void precalculateABC(float4* ABCm, float* M, float timestep, float alpha, unsigned int numPoints)
{
int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

if (me_idx>=numPoints)
return;

float twodelta = timestep*2.0f;
float deltasqr = timestep*timestep;


float Mii = M[me_idx];
float Dii = alpha*Mii;  // mass-proportional damping is applied

//	printf("M: %f\n",Mii);

float Ai = 1.0f/(Dii/twodelta + Mii/deltasqr);
float Bi = ((2.0f*Mii)/deltasqr)*Ai;
float Ci = (Dii/twodelta)*Ai - 0.5f*Bi;

//	printf("ABC for node %i: %f, %f, %f \n", me_idx, Ai, Bi, Ci);

ABCm[me_idx] = make_float4(Ai,Bi,Ci,Mii);
}