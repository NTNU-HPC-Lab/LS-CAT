#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void kinematics_phase2(float* force, float* vel, int len){
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = bx*blockDim.x + tx;
//if (index == 0){ printf("You have been trolled! \n"); }
if (index < len){
vel[index] += 0.5 * force[index] / MASS_ATOM * time_step;
}
}