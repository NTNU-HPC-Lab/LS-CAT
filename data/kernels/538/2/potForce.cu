#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void potForce(float * PairWise, int N, float * PotOut, float * ForceOut)
{
/*
PairWise - PairWise distances between atoms passed from global
N - # atoms
RowSize - # PairWise distances per block
RowCumSize - # nonzero RowSize array elements = # blocks launched in parallel
PotOut - Store the output Potential in global memory
ForceOut - Store the output Force in global memory along x, 1D array size N*N
*/
int bx = blockIdx.x;
int tx = threadIdx.x;
//Register variables to store pairwise separation
float delx;
float dely;
float delz;
float delr2, delrm6;
float Potential;
float Forcex;
float Forcey;
float Forcez;
int row = tx + bx*BLOCK_SIZE;
//if (row == 0) printf("I'm in 1! \n");
if (row < N*N)
{
delx = PairWise[row];
dely = PairWise[row + N*N];
delz = PairWise[row + N*N * 2];
delr2 = delx*delx + dely*dely + delz*delz;
delrm6 = __powf(delr2, (float)-3);
if (delr2 == 0.0) {
Potential = 0;
Forcex = 0;
Forcey = 0;
Forcez = 0;
}
else{
Potential = 4 * __fadd_rn(delrm6*delrm6, -1 * delrm6);
Forcex = -(delx / delr2) * 24 * __fadd_rn(2 * delrm6*delrm6, -1 * delrm6);
Forcey = -(dely / delr2) * 24 * __fadd_rn(2 * delrm6*delrm6, -1 * delrm6);
Forcez = -(delz / delr2) * 24 * __fadd_rn(2 * delrm6*delrm6, -1 * delrm6);
}
PotOut[row] = Potential;
ForceOut[row] = Forcex;
ForceOut[row + N*N] = Forcey;
ForceOut[row + N*N * 2] = Forcez;
}

}