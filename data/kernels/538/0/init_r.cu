#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void init_r(float* r, int N_cube){
int ix = threadIdx.x + blockDim.x* blockIdx.x;
int iy = threadIdx.y + blockDim.y* blockIdx.y;
int iz = threadIdx.z + blockDim.z* blockIdx.z;
int index = ix + iy*N_cube + iz * N_cube * N_cube;
if (ix < N_cube && iy < N_cube && iz<N_cube && index < N_ATOMS){
r[index] = L / 2.0 * (1.0 - float(2 * ix + 1) / N_cube);
r[index + N_ATOMS] = L / 2.0 * (1.0 - float(2 * iy + 1) / N_cube);
r[index + 2 * N_ATOMS] = L / 2.0 * (1.0 - float(2 * iz + 1) / N_cube);
}
}