#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__device__ float PutInBox(float r){
if (fabs(r) > L / 2.0)
r += (2 * (r < 0) - 1)*ceil((fabs(r) - L / 2.0f) / L)*L;
return r;
}
__global__ void kinematics(float* positions, float* force, float* vel, int len){
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = bx*blockDim.x + tx;
float tempr;
//if (index == 0){ printf("You have been trolled! \n"); }
if (index < len){
tempr = positions[index] + 0.5f * force[index] / MASS_ATOM * time_step*time_step + vel[index] * time_step;
positions[index] = PutInBox(tempr);
vel[index] += force[index] / MASS_ATOM * time_step;
}
}