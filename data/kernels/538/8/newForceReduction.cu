#include "hip/hip_runtime.h"
#include "includes.h"
// Include files


// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void newForceReduction(float *input, float *output, int startunit, int len)
{
unsigned int tx = threadIdx.x;
unsigned int start = blockIdx.x *N_ATOMS;

__shared__ float partSum[BLOCK_SIZE];
// if (tx == 0) printf("Length of the shared memory array - %i \n", N_ATOMS);

//Loading input floats to shared memory
//Take care of the boundary conditions
if (tx < N_ATOMS) { partSum[tx] = input[start + tx]; }
else{ partSum[tx] = 0.0f; }

__syncthreads();

//Reduction Kernel for each dimension
if (tx < 512){
partSum[tx] += partSum[tx + 512];
} __syncthreads();
if (tx < 256){
partSum[tx] += partSum[tx + 256];
} __syncthreads();
if (tx < 128){
partSum[tx] += partSum[tx + 128];
} __syncthreads();
if (tx < 64){
partSum[tx] += partSum[tx + 64];
} __syncthreads();
if (tx < 32){
partSum[tx] += partSum[tx + 32];
partSum[tx] += partSum[tx + 16];
partSum[tx] += partSum[tx + 8];
partSum[tx] += partSum[tx + 4];
partSum[tx] += partSum[tx + 2];
partSum[tx] += partSum[tx + 1];
}
if (tx == 0){
output[blockIdx.x] = -partSum[0];
}
}