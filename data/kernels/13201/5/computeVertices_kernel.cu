#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeVertices_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
unsigned int x = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

// calculate uv coordinates
float u = x / (float) width;
float v = y / (float) height;
u = u*2.0 - 1.0f;
v = v*2.0 - 1.0f;

// calculate simple sine wave pattern
float freq = 4.0f;
float w = sin(u*freq + time) * cos(v*freq + time) * 0.5f;

// write output vertex
pos[y*width+x] = make_float4(u, w, v, 1.0f);
}