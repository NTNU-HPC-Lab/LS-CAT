#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX








const unsigned int BLOCK_SIZE = 512;

__global__ void fillAndAddKernelV2(float* c, float *a, float* b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
a[i] = sin((double)i)*sin((double)i);
b[i] = cos((double)i)*cos((double)i);
c[i] = a[i] + b[i];
}