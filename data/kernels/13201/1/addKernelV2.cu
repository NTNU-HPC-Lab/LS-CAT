#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX








const unsigned int BLOCK_SIZE = 512;

__global__ void addKernelV2(float *c, const float *a, const float *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = a[i] + b[i];
}