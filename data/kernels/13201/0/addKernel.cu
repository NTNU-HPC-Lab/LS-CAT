#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX








const unsigned int BLOCK_SIZE = 512;

__global__ void addKernel(float *c, const float *a, const float *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}