#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX








const unsigned int BLOCK_SIZE = 512;

__global__ void fillKernelV2(float *a, float* b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
a[i] = sin((double)i)*sin((double)i);
b[i] = cos((double)i)*cos((double)i);
}