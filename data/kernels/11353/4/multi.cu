#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multi(int *a, int *b, int *c,int n) {
int suma = 0;
int row = blockIdx.y * blockDim.y + threadIdx.y ;
int col = blockIdx.x * blockDim.x + threadIdx.x ;

if (row <n && col<n){
for(int i=0;i<N;++i){
suma+= a[row*n+i] * b[i*n+col];
}
}
c[row*n+col] = suma;
}