#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(float * d_out, float * d_in) {
const unsigned int lid = threadIdx.x;
const unsigned int gid = blockIdx.x*blockDim.x + lid;
float f = d_in[gid];
d_out[gid] = f * f;
}