#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void special(float * d_out, float * d_in, int size) {
const unsigned int lid = threadIdx.x;
const unsigned int gid = blockIdx.x*blockDim.x + lid;
if(gid < size) {
float x = d_in[gid];
d_out[gid] = powf(x / (x - 2.3), 3);
}
}