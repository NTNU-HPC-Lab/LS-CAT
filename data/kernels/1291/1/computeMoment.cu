#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMoment(int *readArr, int *writeArr, double *weightArr, int n){
int row = blockIdx.x*blockDim.x + threadIdx.x;
int col = blockIdx.y*blockDim.y + threadIdx.y;

// If coordinates are between boundaries
// update the write array accordingly
if(row < 517 && col < 517){
float influence = 0.0f;
for (int i=-2; i<3; i++)
{
for (int j=-2; j<3; j++)
{
//add extra n so that modulo behaves like mathematics modulo
//that is return only positive values
int y = (row+i+n)%n;
int x = (col+j+n)%n;
influence += weightArr[i*5 + j]*readArr[y*n + x];
}
}

writeArr[row*n + col] = readArr[row*n + col];
if 		(influence<-diff)	writeArr[row*n + col] = -1;
else if (influence>diff)	writeArr[row*n + col] = 1;
}
__syncthreads();

}