#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMoment(int *readArr, int *writeArr, double *weightArr, int n){
// The dimensions are hardcoded here to simplify extra syntax
// cuda uses for dynamic shared memory allocation
__shared__ int readArr_shared[32][32];
__shared__ double weightArr_shared[5][5];

int row = blockIdx.x*blockDim.x + threadIdx.x;
int col = blockIdx.y*blockDim.y + threadIdx.y;

if(threadIdx.x<5 && threadIdx.y < 5){
weightArr_shared[threadIdx.x][threadIdx.y] = weightArr[threadIdx.x*WINDOW_SIZE + threadIdx.y];
}
__syncthreads();

// Only values within the below borders will be used but the __syncthreads()
// function has to be called outside if statements so we load everything here
readArr_shared[threadIdx.x][threadIdx.y] = readArr[row*n + col];
__syncthreads();

// If coordinates are between boundaries
// update the write array accordingly
if(row < n && col < n){
float influence = 0.0f;
for (int i=-2; i<3; i++)
{
for (int j=-2; j<3; j++)
{
//add extra n so that modulo behaves like mathematics modulo
//that is return only positive values
if(threadIdx.x >= MIN_MARGIN && threadIdx.y >= MIN_MARGIN &&
threadIdx.x <= 31-MIN_MARGIN && threadIdx.y <= 31-MIN_MARGIN){
int y = threadIdx.x + i;
int x = threadIdx.y + j;
influence += weightArr_shared[i+2][j+2]*readArr_shared[y][x];
}else{
int y = (row+i+n)%n;
int x = (col+j+n)%n;
influence += weightArr_shared[i+2][j+2]*readArr[y*n + x];
}
}
}

if(threadIdx.x >= MIN_MARGIN && threadIdx.y >= MIN_MARGIN &&
threadIdx.x <= 31-MIN_MARGIN && threadIdx.y <= 31-MIN_MARGIN){
writeArr[row*n + col] = readArr_shared[threadIdx.x][threadIdx.y];
if 		(influence<-diff)	writeArr[row*n + col] = -1;
else if (influence>diff)	writeArr[row*n + col] = 1;
}else {
writeArr[row*n + col] = readArr[row*n + col];
if 		(influence<-diff)	writeArr[row*n + col] = -1;
else if (influence>diff)	writeArr[row*n + col] = 1;
}
}
__syncthreads();

}