#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMoment(int8_t *readArr, int8_t *writeArr, float *weightArr, int n, int tileSize){
int row_init = blockIdx.x*(blockDim.x*tileSize) + threadIdx.x*tileSize;
int col_init = blockIdx.y*(blockDim.y*tileSize) + threadIdx.y*tileSize;

// Assign each thread a tileSizeXtileSize tile
for(int ii=0; ii<tileSize; ++ii){
for (int jj=0; jj<tileSize; ++jj){
int row = row_init + ii;
int col = col_init + jj;

// If coordinates are between boundaries
// update the write array accordingly
if(row < n && col < n){
float influence = 0.0f;
for (int i=-2; i<3; i++)
{
for (int j=-2; j<3; j++)
{
//add extra n so that modulo behaves like mathematics modulo
//that is return only positive values
int y = (row+i+n)%n;
int x = (col+j+n)%n;
influence += weightArr[i*5 + j]*readArr[y*n + x];
}
}

writeArr[row*n + col] = readArr[row*n + col];
if 	(influence<-diff)	writeArr[row*n + col] = -1;
else if (influence>diff)	writeArr[row*n + col] = 1;
__syncthreads();
}
}
}
}