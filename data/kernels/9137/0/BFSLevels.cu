#include "hip/hip_runtime.h"
#include "includes.h"
// Contains GPU Cuda code that executes BFS algorithm

// STL

// Internal Headers

// taken from global_memory.cu, Creates event and records time
__global__ void BFSLevels(int  *vertices, int  *edges, int  *distances, int  *predecessors, int  *vertIndices, int  *edgeSize, bool *levels, bool *visitedVertices, bool *foundDest, int   numVert, int   destination)
{
// Grab ThreadID
int thrID = threadIdx.x + blockIdx.x * blockDim.x;

__shared__ bool destFound;
destFound = false;

if (thrID < numVert && !destFound)
{
int curVert = vertices[thrID];

// Iterate through level if true
if (levels[curVert])
{
levels[curVert]          = false;
visitedVertices[curVert] = true;

// Grab indexes for curVert edges in edge array
int edgesEnd  = edgeSize[thrID] + vertIndices[thrID];

// Iterate through all edges for current vertex
for (int edgeIter = vertIndices[thrID]; edgeIter < edgesEnd; ++edgeIter)
{
// Grab next Vertex at end of edge
int nextVert = edges[edgeIter];

// If it hasn't been visited store info
// for distance and predecessors and set level
// to true for next level of vertices
if (!visitedVertices[nextVert])
{
distances[nextVert] = distances[curVert] + 1;
levels[nextVert] = true;
predecessors[nextVert]  = curVert;

// Set found destination to true and sync threads
if (nextVert == destination)
{
*foundDest = true;
destFound  = true;
__syncthreads();
}
}
}
}
}
}