#include "hip/hip_runtime.h"
#include "includes.h"

/*Performs separable convolution on 3d cube*/



__global__ void convolution_sep(float *output, const float *input, const float *kernel, const int kernel_size, const dim3 imsize, int dir)
{
size_t ix, iy, iz;
if (dir == X_DIR)
{
ix = blockDim.x*blockIdx.x + threadIdx.x;
iy = blockDim.y*blockIdx.y + threadIdx.y;
iz = blockIdx.z;
}
else if (dir == Y_DIR)
{
iy = blockDim.x*blockIdx.x + threadIdx.x;
ix = blockDim.y*blockIdx.y + threadIdx.y;
iz = blockIdx.z;
}
else if (dir == EPS_DIR)
{
iz = blockDim.x*blockIdx.x + threadIdx.x;
ix = blockDim.y*blockIdx.y + threadIdx.y;
iy = blockIdx.z;
}

const bool valid = ix < imsize.x && iy < imsize.y && iz < imsize.z;
const size_t cube_idx = ix + iy*imsize.x + iz*imsize.x*imsize.y;

const size_t radius_size = kernel_size / 2;

extern __shared__ float s_image[]; //size is on kernel call
const size_t s_dim_x = blockDim.x + 2 * radius_size;
const size_t s_ix = radius_size + threadIdx.x;
const size_t s_iy = threadIdx.y;
float result = 0.0;

if (threadIdx.x < radius_size) //is on the left part of the shared memory
{
s_image[s_ix - radius_size + s_iy*s_dim_x] = 0.0f;
}
if (threadIdx.x >= (blockDim.x - radius_size)) //is on the right part
{
s_image[s_ix + radius_size + s_iy*s_dim_x] = 0.0f;
}



s_image[s_ix + s_iy*s_dim_x] = (valid) ? input[cube_idx] : 0.0f;


__syncthreads();


#pragma unroll
for (int i = 0; i < kernel_size; i++)
{
result += kernel[i] * s_image[s_ix - i + radius_size + s_iy*s_dim_x];
}

if (valid)
{

output[cube_idx] = result;
}
}