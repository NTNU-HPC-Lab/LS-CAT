#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cubefilling_atomic(const float* image, float *dev_cube_wi, float *dev_cube_w, const dim3 image_size, int scale_xy, int scale_eps, dim3 dimensions_down)
{
const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
const size_t j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < image_size.x && j < image_size.y) {
const float k = image[i + image_size.x*j];
const size_t cube_idx = (i / scale_xy) + dimensions_down.x*(j / scale_xy) + dimensions_down.x*dimensions_down.y*((int)k / scale_eps);


atomicAdd(&dev_cube_wi[cube_idx], k);
atomicAdd(&dev_cube_w[cube_idx], 1.0f);

}


}