#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cubefilling_loop(const float* image, float *dev_cube_wi, float *dev_cube_w, const dim3 image_size, int scale_xy, int scale_eps, dim3 dimensions_down)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < dimensions_down.x && j < dimensions_down.y) {

size_t cube_idx_1 = i + dimensions_down.x*j;
#pragma unroll
for (int ii = 0; ii < scale_xy; ii++)
{
#pragma unroll
for (int jj = 0; jj < scale_xy; jj++)
{
size_t i_idx = scale_xy*i + ii;
size_t j_idx = scale_xy*j + jj;
if (i_idx < image_size.x && j_idx < image_size.y)
{

float k = image[i_idx + image_size.x*j_idx];
size_t cube_idx_2 = cube_idx_1 + dimensions_down.x*dimensions_down.y*floorf(k / scale_eps);
dev_cube_wi[cube_idx_2] += k;
dev_cube_w[cube_idx_2] += 1.0f;
}

}
}
}


}