#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000


enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void extract_boundary_kernel(float *maskIn, int *boundryPixelArray, int source_nchannel, int source_width, int source_height){
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
for(int channel = 0; channel < source_nchannel; channel++){
if(x<source_width && y<source_height){
int id = x + source_width * y + source_width * source_height * channel;
if(x==0 && y==0 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x==0 && y==source_height-1 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x==source_width-1 && y==0 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x==source_width-1 && y==source_height-1 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x==0 && y < source_height-1 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x==source_width-1 && y < source_height-1 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x < source_width-1 && y==0 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else if(x < source_width-1 && y==source_height-1 && maskIn[id]){
boundryPixelArray[id]=OUTSIDE;
}
else{
int id_right = x+1 + y*source_width + channel * source_width * source_height;
int id_left = x-1 + y*source_width + channel * source_width * source_height;
int id_up = x + (y+1)*source_width + channel * source_width * source_height;
int id_down = x + (y-1)*source_width + channel * source_width * source_height;

if(maskIn[id]>=0.5&&maskIn[id_right]>=0.5&&maskIn[id_left]>=0.5&&maskIn[id_up]>=0.5&&maskIn[id_down]>=0.5){
boundryPixelArray[id] = INSIDE_MASK;
}
else if(maskIn[id]){
boundryPixelArray[id] = BOUNDRY;
}
else{
boundryPixelArray[id] = OUTSIDE;
}
}
}
}
}