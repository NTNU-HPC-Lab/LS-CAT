#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000


enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void poisson_jacobi_kernel(float *targetimg, float *outimg, int *boundary_array,int c, int w, int h, int boundBoxMinX, int boundBoxMaxX, int boundBoxMinY, int boundBoxMaxY){

int x = threadIdx.x + blockIdx.x * blockDim.x + boundBoxMinX;
int y = threadIdx.y + blockIdx.y * blockDim.y + boundBoxMinY;
for(int channel = 0; channel < c; channel++){
int id = x + y*w + channel * w * h;
int idx_nextX = x+1 + w*y +w*h*channel;
int idx_prevX = x-1 + w*y + w*h*channel;
int idx_nextY = x + w*(y+1) +w*h*channel;
int idx_prevY = x + w*(y-1) +w*h*channel;
//printf("id: %d, idx_nextX: %d, idx_prevX: %d, idx_nextY: %d, idx_prevY: %d\n", id, idx_nextX, idx_prevX, idx_nextY, idx_prevY);
if(boundary_array[id] == INSIDE_MASK){
double neighbor_target = targetimg[idx_nextY]+targetimg[idx_nextX]+targetimg[idx_prevX]+targetimg[idx_prevY];
double neighbor_output = outimg[idx_nextY]+outimg[idx_nextX]+outimg[idx_prevX]+outimg[idx_prevY];
outimg[id] = 0.25*(4*targetimg[id]-neighbor_target + neighbor_output);
}
}

}