#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000


enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void merge_without_blend_kernel(float *srcimg, float *targetimg, float *outimg, int *boundary_array,int source_nchannel, int source_width, int source_height){
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
for(int channel = 0; channel < source_nchannel; channel++){
int id = x + y*source_width + channel * source_width * source_height;
if(boundary_array[id] == INSIDE_MASK){
outimg[id] = targetimg[id];
}
else{
outimg[id] = srcimg[id];
}
}
}