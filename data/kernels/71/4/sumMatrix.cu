#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrix(float *A, float *B, float *C, int nx, int ny) {
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
unsigned int idx = ix + iy * nx;
if(ix < nx && iy < ny) {
C[idx] = A[idx] + B[idx];
}
}