#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
int id = threadIdx.x;
C[id] = A[id] + B[id];
}