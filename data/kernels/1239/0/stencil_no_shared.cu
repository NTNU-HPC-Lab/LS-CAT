#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE 1024

#ifndef RADIUS
#define RADIUS 3
#endif

#ifndef ITERS
#define ITERS 100
#endif

#ifndef USE_L2
#define USE_L2 false
#endif





__global__ void stencil_no_shared(int *in, int *out)
{
int temp[BLOCK_SIZE + 2 * RADIUS];
int gindex = threadIdx.x + blockIdx.x * blockDim.x;
int lindex = threadIdx.x + RADIUS;
// Read input elements into shared memory
temp[lindex] = in[gindex+RADIUS];
if (threadIdx.x < RADIUS) {
temp[lindex - RADIUS] = in[gindex];
temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
}
__syncthreads();
// Apply the stencil
int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
result += temp[lindex + offset];
// Store the result
out[gindex] = result;

}