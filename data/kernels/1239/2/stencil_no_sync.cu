#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_no_sync(int *in, int *out)
{
__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
int gindex = threadIdx.x + blockIdx.x * blockDim.x;
int lindex = threadIdx.x + RADIUS;
// Read input elements into shared memory
temp[lindex] = in[gindex+RADIUS];
if (threadIdx.x < RADIUS) {
temp[lindex - RADIUS] = in[gindex];
temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
}
////////////////////////////// missing sync thread ////////////////////////

// Apply the stencil
int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
result += temp[lindex + offset];
// Store the result
out[gindex] = result;

}