#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {

int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
int p;

float2 xterm, yterm;
// gtidx is the domain location in x for this thread
if (gtidx < dx) {
for (p = 0; p < lb; p++) {
// fi is the domain location in y for this thread
int fi = gtidy + p;
if (fi < dy) {
int fj = fi * dx + gtidx;
xterm = vx[fj];
yterm = vy[fj];

// Compute the index of the wavenumber based on the
// data order produced by a standard NN FFT.
int iix = gtidx;
int iiy = (fi>dy/2)?(fi-(dy)):fi;

// Velocity diffusion
float kk = (float)(iix * iix + iiy * iiy); // k^2
float diff = 1.f / (1.f + visc * dt * kk);
xterm.x *= diff; xterm.y *= diff;
yterm.x *= diff; yterm.y *= diff;

// Velocity projection
if (kk > 0.f) {
float rkk = 1.f / kk;
// Real portion of velocity projection
float rkp = (iix * xterm.x + iiy * yterm.x);
// Imaginary portion of velocity projection
float ikp = (iix * xterm.y + iiy * yterm.y);
xterm.x -= rkk * rkp * iix;
xterm.y -= rkk * ikp * iix;
yterm.x -= rkk * rkp * iiy;
yterm.y -= rkk * ikp * iiy;
}

vx[fj] = xterm;
vy[fj] = yterm;
}
}
}
}