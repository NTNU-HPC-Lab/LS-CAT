#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];


__global__ void int2lin_resmpl_messy_gpu_kernel(float *dev_in_img, float *dev_out_img, float *dev_C0_tmp, float *dev_C1_tmp, float *dev_C2_tmp, int org_wd, int org_ht, int dst_wd, int dst_ht, int n_channels, float r, int hn, int wn, int xbd0, int xbd1, int ybd0, int ybd1, int *xas_const, int *xbs_const, float *xwts_const, int *yas_const, int *ybs_const, float *ywts_const)
{

unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

if ((x_pos < dst_wd) && (y_pos < dst_ht)) {

int xa, ya, yb;
float wt, wt1;
float *A00, *A01, *A02, *A03, *B00;
float *A10, *A11, *A12, *A13, *B10;
float *A20, *A21, *A22, *A23, *B20;

float *A0 = dev_in_img + 0;
float *B0 = dev_out_img + (0 * dst_ht * dst_wd);
float *A1 = dev_in_img + 1;
float *B1 = dev_out_img + (1 * dst_ht * dst_wd);
float *A2 = dev_in_img + 2;
float *B2 = dev_out_img + (2 * dst_ht * dst_wd);
int y1 = 0;

if (org_ht > dst_ht) {
int m = 1;
for (int iter = 0; iter < y_pos; iter++) {
while (y1 + m < hn && yb == ybs_const[y1 + m])
m++;
y1 += m;
}
wt = ywts_const[y1];
wt1 = 1 - wt;
} else {
y1 = y_pos;
wt = ywts_const[y1];
wt1 = 1 - wt;
}

if (y_pos == 0)
y1 = 0;

ya = yas_const[y1];
A00 = A0 + (ya * org_wd * n_channels);
A01 = A00 + (org_wd * n_channels);
A02 = A01 + (org_wd * n_channels);
A03 = A02 + (org_wd * n_channels);

A10 = A1 + (ya * org_wd * n_channels);
A11 = A00 + (org_wd * n_channels);
A12 = A01 + (org_wd * n_channels);
A13 = A02 + (org_wd * n_channels);

A20 = A2 + (ya * org_wd * n_channels);
A21 = A00 + (org_wd * n_channels);
A22 = A01 + (org_wd * n_channels);
A23 = A02 + (org_wd * n_channels);

yb = ybs_const[y1];
B00 = B0 + (yb * dst_wd);
B10 = B1 + (yb * dst_wd);
B20 = B2 + (yb * dst_wd);

if (x_pos < org_wd) {

// resample along y direction
if (org_ht > dst_ht) {
int m = 1;
while ((y1 + m < hn) && (yb == ybs_const[y1 + m]))
m++;

if (m == 1) {
dev_C0_tmp[x_pos] = A00[x_pos * n_channels] * ywts_const[y1];
dev_C1_tmp[x_pos] = A10[x_pos * n_channels] * ywts_const[y1];
dev_C2_tmp[x_pos] = A20[x_pos * n_channels] * ywts_const[y1];
} else if (m == 2) {
dev_C0_tmp[x_pos] = (A00[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A01[x_pos * n_channels] * ywts_const[y1 + 1]);
dev_C1_tmp[x_pos] = (A10[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A11[x_pos * n_channels] * ywts_const[y1 + 1]);
dev_C2_tmp[x_pos] = (A20[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A21[x_pos * n_channels] * ywts_const[y1 + 1]);
} else if (m == 3) {
dev_C0_tmp[x_pos] = (A00[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A01[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A02[x_pos * n_channels] * ywts_const[y1 + 2]);
dev_C1_tmp[x_pos] = (A10[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A11[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A12[x_pos * n_channels] * ywts_const[y1 + 2]);
dev_C2_tmp[x_pos] = (A20[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A21[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A22[x_pos * n_channels] * ywts_const[y1 + 2]);
} else if (m >= 4) {
dev_C0_tmp[x_pos] = (A00[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A01[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A02[x_pos * n_channels] * ywts_const[y1 + 2]) +
(A03[x_pos * n_channels] * ywts_const[y1 + 3]);
dev_C1_tmp[x_pos] = (A10[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A11[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A12[x_pos * n_channels] * ywts_const[y1 + 2]) +
(A13[x_pos * n_channels] * ywts_const[y1 + 3]);
dev_C2_tmp[x_pos] = (A20[x_pos * n_channels] * ywts_const[y1 + 0]) +
(A21[x_pos * n_channels] * ywts_const[y1 + 1]) +
(A22[x_pos * n_channels] * ywts_const[y1 + 2]) +
(A23[x_pos * n_channels] * ywts_const[y1 + 3]);
}

for (int y0 = 4; y0 < m; y0++) {
A01 = A00 + (y0 * org_wd * n_channels);
A11 = A10 + (y0 * org_wd * n_channels);
A11 = A10 + (y0 * org_wd * n_channels);
wt1 = ywts_const[y1 + y0];
dev_C0_tmp[x_pos] = dev_C0_tmp[x_pos] + (A01[x_pos * n_channels] * wt1);
dev_C1_tmp[x_pos] = dev_C1_tmp[x_pos] + (A11[x_pos * n_channels] * wt1);
dev_C2_tmp[x_pos] = dev_C2_tmp[x_pos] + (A21[x_pos * n_channels] * wt1);
}

} else {
bool yBd = y_pos < ybd0 || y_pos >= dst_ht - ybd1;

if (yBd) {
dev_C0_tmp[x_pos] = A00[x_pos * n_channels];
dev_C1_tmp[x_pos] = A10[x_pos * n_channels];
dev_C2_tmp[x_pos] = A20[x_pos * n_channels];
} else {
dev_C0_tmp[x_pos] = (A00[x_pos * n_channels] * wt) + (A01[x_pos * n_channels] * wt1);
dev_C1_tmp[x_pos] = (A10[x_pos * n_channels] * wt) + (A11[x_pos * n_channels] * wt1);
dev_C2_tmp[x_pos] = (A20[x_pos * n_channels] * wt) + (A21[x_pos * n_channels] * wt1);
}
}
}

/* ensure that all threads have calculated the values for C until this point */
__syncthreads();

if (x_pos < dst_wd) {
// resample along x direction (B -> C)
if (org_wd > dst_wd) {
if (xbd0 == 2) {
xa = xas_const[x_pos * 4];
B00[x_pos] = (dev_C0_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) + (dev_C0_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]);
B10[x_pos] = (dev_C1_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) + (dev_C1_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]);
B20[x_pos] = (dev_C2_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) + (dev_C2_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]);
} else if (xbd0 == 3) {
xa = xas_const[x_pos * 4];
B00[x_pos] = (dev_C0_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C0_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C0_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]);
B10[x_pos] = (dev_C1_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C1_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C1_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]);
B20[x_pos] = (dev_C2_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C2_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C2_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]);

} else if (xbd0 == 4) {
xa = xas_const[x_pos * 4];
B00[x_pos] = (dev_C0_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C0_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C0_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]) +
(dev_C0_tmp[xa + 3] * xwts_const[(4 * x_pos) + 3]);
B10[x_pos] = (dev_C1_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C1_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C1_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]) +
(dev_C1_tmp[xa + 3] * xwts_const[(4 * x_pos) + 3]);
B20[x_pos] = (dev_C2_tmp[xa + 0] * xwts_const[(4 * x_pos) + 0]) +
(dev_C2_tmp[xa + 1] * xwts_const[(4 * x_pos) + 1]) +
(dev_C2_tmp[xa + 2] * xwts_const[(4 * x_pos) + 2]) +
(dev_C2_tmp[xa + 3] * xwts_const[(4 * x_pos) + 3]);

} else if (xbd0 > 4) {
for(int x = 0; x < wn; x++) {
B00[xbs_const[x]] += dev_C0_tmp[xas_const[x]] * xwts_const[x];
B10[xbs_const[x]] += dev_C1_tmp[xas_const[x]] * xwts_const[x];
B20[xbs_const[x]] += dev_C2_tmp[xas_const[x]] * xwts_const[x];
}
}
} else {
int x = 0;
for (x = 0; x < xbd0; x++) {
B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x];
B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x];
B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x];
}
for (; x < dst_wd - xbd1; x++) {
B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x] + dev_C0_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x] + dev_C1_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x] + dev_C2_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
}
for (; x < dst_wd; x++) {
B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x];
B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x];
B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x];
}
}
}

__syncthreads();
}
}