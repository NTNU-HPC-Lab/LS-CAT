#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];


__global__ void convert_to_luv_gpu_kernel(unsigned char *in_img, float *out_img, int cols, int rows, bool use_rgb)
{
float r, g, b, l, u, v, x, y, z, lt;

unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

if ((x_pos < cols) && (y_pos < rows)) {

unsigned int pos = (y_pos * cols) + x_pos;

if (use_rgb) {
r = (float)in_img[(3 * pos)];
g = (float)in_img[(3 * pos) + 1];
b = (float)in_img[(3 * pos) + 2];
} else {
b = (float)in_img[(3 * pos)];
g = (float)in_img[(3 * pos) + 1];
r = (float)in_img[(3 * pos) + 2];
}

x = (mr_const[0] * r) + (mg_const[0] * g) + (mb_const[0] * b);
y = (mr_const[1] * r) + (mg_const[1] * g) + (mb_const[1] * b);
z = (mr_const[2] * r) + (mg_const[2] * g) + (mb_const[2] * b);

float maxi = 1.0f / 270;
float minu = -88.0f * maxi;
float minv = -134.0f * maxi;
float un = 0.197833f;
float vn = 0.468331f;

lt = lTable_const[static_cast<int>((y*1024))];
l = lt; z = 1/(x + (15 * y) + (3 * z) + (float)1e-35);
u = lt * (13 * 4 * x * z - 13 * un) - minu;
v = lt * (13 * 9 * y * z - 13 * vn) - minv;

out_img[(3 * pos)] = l;
out_img[(3 * pos) + 1] = u;
out_img[(3 * pos) + 2] = v;
}
}