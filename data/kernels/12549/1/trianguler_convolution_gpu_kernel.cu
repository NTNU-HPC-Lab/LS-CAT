#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];


__global__ void trianguler_convolution_gpu_kernel(float *dev_I, float *dev_O, float *T0, float *T1, float *T2, int wd, int ht, float nrm, float p)
{
unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

if ((x_pos < wd) && (y_pos < ht)) {

float *It0, *It1, *It2, *Im0, *Im1, *Im2, *Ib0, *Ib1, *Ib2;
float *Ot0, *Ot1, *Ot2;
float *T00, *T10, *T20;


It0 = Im0 = Ib0 = dev_I + (y_pos * wd) + (0 * ht * wd);
It1 = Im1 = Ib1 = dev_I + (y_pos * wd) + (1 * ht * wd);
It2 = Im2 = Ib2 = dev_I + (y_pos * wd) + (2 * ht * wd);

Ot0 = dev_O + (y_pos * wd) + (0 * ht * wd);
Ot1 = dev_O + (y_pos * wd) + (1 * ht * wd);
Ot2 = dev_O + (y_pos * wd) + (2 * ht * wd);

T00 = T0 + (y_pos * wd);
T10 = T1 + (y_pos * wd);
T20 = T2 + (y_pos * wd);

if(y_pos > 0) { /// not the first row, let It point to previous row
It0 -= wd;
It1 -= wd;
It2 -= wd;
}
if(y_pos < ht - 1) { /// not the last row, let Ib point to next row
Ib0 += wd;
Ib1 += wd;
Ib2 += wd;
}

T00[x_pos] = nrm * (It0[x_pos] + (p * Im0[x_pos]) + Ib0[x_pos]);
T10[x_pos] = nrm * (It1[x_pos] + (p * Im1[x_pos]) + Ib1[x_pos]);
T20[x_pos] = nrm * (It2[x_pos] + (p * Im2[x_pos]) + Ib2[x_pos]);

__syncthreads();

if (x_pos == 0) {
Ot0[x_pos] = ((1 + p) * T00[x_pos]) + T00[x_pos + 1];
Ot1[x_pos] = ((1 + p) * T10[x_pos]) + T10[x_pos + 1];
Ot2[x_pos] = ((1 + p) * T20[x_pos]) + T20[x_pos + 1];
} else if (x_pos == wd - 1) {
Ot0[x_pos] = T00[x_pos - 1] + ((1 + p) * T00[x_pos]);
Ot1[x_pos] = T10[x_pos - 1] + ((1 + p) * T10[x_pos]);
Ot2[x_pos] = T20[x_pos - 1] + ((1 + p) * T20[x_pos]);
} else {
Ot0[x_pos] = T00[x_pos - 1] + (p * T00[x_pos]) + T00[x_pos + 1];
Ot1[x_pos] = T10[x_pos - 1] + (p * T10[x_pos]) + T10[x_pos + 1];
Ot2[x_pos] = T20[x_pos - 1] + (p * T20[x_pos]) + T20[x_pos + 1];
}

__syncthreads();
}

}