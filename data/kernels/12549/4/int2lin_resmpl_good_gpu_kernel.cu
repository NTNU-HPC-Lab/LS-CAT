#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];


__global__ void int2lin_resmpl_good_gpu_kernel(float *dev_in_img, float *dev_out_img, float *dev_C0_tmp, float *dev_C1_tmp, float *dev_C2_tmp, int org_wd, int org_ht, int dst_wd, int dst_ht, int n_channels, float r, int *yas_const, int *ybs_const)
{

unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

if ((x_pos < dst_wd) && (y_pos < dst_ht)) {

int ya, yb;
float *A00, *A01, *A02, *A03, *B00;
float *A10, *A11, *A12, *A13, *B10;
float *A20, *A21, *A22, *A23, *B20;

float *A0 = dev_in_img + 0;
float *B0 = dev_out_img + (0 * dst_ht * dst_wd);
float *A1 = dev_in_img + 1;
float *B1 = dev_out_img + (1 * dst_ht * dst_wd);
float *A2 = dev_in_img + 2;
float *B2 = dev_out_img + (2 * dst_ht * dst_wd);

if (org_ht == dst_ht && org_wd == dst_wd) {
int out_img_idx = y_pos + (dst_wd * x_pos);
B0[out_img_idx] = A0[out_img_idx * n_channels];
B1[out_img_idx] = A1[out_img_idx * n_channels];
B2[out_img_idx] = A2[out_img_idx * n_channels];
return;
}

int y1 = 0;

if (org_ht == 2 * dst_ht) {
y1 += 2 * y_pos;
} else if (org_ht == 3 * dst_ht) {
y1 += 3 * y_pos;
} else if (org_ht == 4 * dst_ht) {
y1 += 4 * y_pos;
}

if (y_pos == 0)
y1 = 0;

ya = yas_const[y1];
A00 = A0 + (ya * org_wd * n_channels);
A01 = A00 + (org_wd * n_channels);
A02 = A01 + (org_wd * n_channels);
A03 = A02 + (org_wd * n_channels);

A10 = A1 + (ya * org_wd * n_channels);
A11 = A00 + (org_wd * n_channels);
A12 = A01 + (org_wd * n_channels);
A13 = A02 + (org_wd * n_channels);

A20 = A2 + (ya * org_wd * n_channels);
A21 = A00 + (org_wd * n_channels);
A22 = A01 + (org_wd * n_channels);
A23 = A02 + (org_wd * n_channels);

yb = ybs_const[y1];
B00 = B0 + (yb * dst_wd);
B10 = B1 + (yb * dst_wd);
B20 = B2 + (yb * dst_wd);

// resample along y direction
if (org_ht == 2 * dst_ht) {
dev_C0_tmp[x_pos] = A00[x_pos * n_channels] + A01[x_pos * n_channels];
dev_C1_tmp[x_pos] = A10[x_pos * n_channels] + A11[x_pos * n_channels];
dev_C2_tmp[x_pos] = A20[x_pos * n_channels] + A21[x_pos * n_channels];
} else if (org_ht == 3 * dst_ht) {
dev_C0_tmp[x_pos] = A00[x_pos * n_channels] + A01[x_pos * n_channels] + A02[x_pos * n_channels];
dev_C1_tmp[x_pos] = A10[x_pos * n_channels] + A11[x_pos * n_channels] + A12[x_pos * n_channels];
dev_C2_tmp[x_pos] = A20[x_pos * n_channels] + A21[x_pos * n_channels] + A22[x_pos * n_channels];
} else if (org_ht == 4 * dst_ht) {
dev_C0_tmp[x_pos] = A00[x_pos * n_channels] + A01[x_pos * n_channels] + A02[x_pos * n_channels] + A03[x_pos * n_channels];
dev_C1_tmp[x_pos] = A10[x_pos * n_channels] + A11[x_pos * n_channels] + A12[x_pos * n_channels] + A13[x_pos * n_channels];
dev_C2_tmp[x_pos] = A20[x_pos * n_channels] + A21[x_pos * n_channels] + A22[x_pos * n_channels] + A23[x_pos * n_channels];
}

/* ensure that all threads have calculated the values for C until this point */
__syncthreads();

// resample along x direction (B -> C)
if (org_wd == 2 * dst_wd) {
B00[x_pos]= (dev_C0_tmp[2 * x_pos] + dev_C0_tmp[(2 * x_pos) + 1]) * (r / 2);
B10[x_pos]= (dev_C1_tmp[2 * x_pos] + dev_C1_tmp[(2 * x_pos) + 1]) * (r / 2);
B20[x_pos]= (dev_C2_tmp[2 * x_pos] + dev_C2_tmp[(2 * x_pos) + 1]) * (r / 2);
} else if (org_wd == 3 * dst_wd) {
B00[x_pos] = (dev_C0_tmp[3 * x_pos] + dev_C0_tmp[(3 * x_pos) + 1] + dev_C0_tmp[(3 * x_pos) + 2]) * (r / 3);
B10[x_pos] = (dev_C1_tmp[3 * x_pos] + dev_C1_tmp[(3 * x_pos) + 1] + dev_C1_tmp[(3 * x_pos) + 2]) * (r / 3);
B20[x_pos] = (dev_C2_tmp[3 * x_pos] + dev_C2_tmp[(3 * x_pos) + 1] + dev_C2_tmp[(3 * x_pos) + 2]) * (r / 3);
} else if (org_wd == 4 * dst_wd) {
B00[x_pos] = (dev_C0_tmp[4 * x_pos] + dev_C0_tmp[(4 * x_pos) + 1] + dev_C0_tmp[(4 * x_pos) + 2] + dev_C0_tmp[(4 * x_pos) + 3]) * (r / 4);
B10[x_pos] = (dev_C1_tmp[4 * x_pos] + dev_C1_tmp[(4 * x_pos) + 1] + dev_C1_tmp[(4 * x_pos) + 2] + dev_C1_tmp[(4 * x_pos) + 3]) * (r / 4);
B20[x_pos] = (dev_C2_tmp[4 * x_pos] + dev_C2_tmp[(4 * x_pos) + 1] + dev_C2_tmp[(4 * x_pos) + 2] + dev_C2_tmp[(4 * x_pos) + 3]) * (r / 4);
}

__syncthreads();
}
}