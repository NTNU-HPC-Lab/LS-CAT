#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_MM_OPT( float *a, float *b, float *c, int wA, int wB, int hA)
{
#define blockTile 16
/* Blocksize is 16x16 */
/* Allocate shared memory */
__shared__ float aBlock[blockTile][blockTile];
__shared__ float bBlock[blockTile][blockTile];

/* Calculate global index X, Y */

int bx = blockIdx.x;
int by = blockIdx.y;

int tx = threadIdx.x;
int ty = threadIdx.y;

int gx = blockDim.x * bx + tx;  // column
int gy = blockDim.y * by + ty;   // row

/* Compute offset idx for A & B */
// First A index (row shift) Block.row * Block.width * A.width
int a0 = wA * 16 * by;
// aBegin -> last element in row -> + width - 1
int aZ = a0 + wA - 1;
// Column block iteration = blockDim.x
int aD = 16;
// b_0 -> Column Shift
int b0 = 16 * bx;
// Row block iteration = blockDim.y * width B
int bD = 16 * wB;

float sum = 0.f;

for(int aI = a0, bI = b0; aI <= aZ; aI += aD, bI += bD)
{

/* Assign shared memory and sync  */
/* Warning, wA*gidy may be out of bounds */
aBlock[ty][tx] = a[aI + ty*wA + tx];
bBlock[ty][tx] = b[bI + ty*wB + tx];

/* Make sure all of the threads have updated the memory cache */
__syncthreads();

/* Sum over NK */
for(int k=0; k < 16; k++)
{
/* C = (A x B) */
sum += aBlock[ty][k] * bBlock[k][tx];
}
}

c[gy*wB + gx] = sum;
//c[i * NJ + j] = ALPHA*sum + BETA*c[i * NJ + j];
}