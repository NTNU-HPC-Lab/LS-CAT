#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuMultOpti( int *a, int *b, int *c, int wA, int wB, int hA)
{
#define blockTile 16
/* Blocksize is 16x16 */
/* Allocate shared memory */
__shared__ int aBlock[blockTile][blockTile];
__shared__ int bBlock[blockTile][blockTile];

/* Calculate global index X, Y*/
int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // column
int gidy = blockDim.y * blockIdx.y + threadIdx.y;   // row

/* Assign shared memory and sync  */
/* Warning, wA*gidy may be out of bounds */
aBlock[threadIdx.x][threadIdx.y] = a[gidy*wA + threadIdx.x];
bBlock[threadIdx.x][threadIdx.y] = b[threadIdx.y*wB + gidx];

/* Make sure all of the threads have cached the memory */
__syncthreads();

/* Check if global IDs are within limits */
if(gidx < wB && gidy < hA)
{
int sum = 0;
for(int k=0; k<wA; k++)
{
sum += aBlock[threadIdx.y][k] * bBlock[k][threadIdx.x];
}
// c [gidy][gidx]
c[gidy * wB + gidx] = sum;

}
}