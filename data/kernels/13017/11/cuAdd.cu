#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAdd(int *a,int *b,int *c, int N)
{
// 1D global index
int offset = blockDim.x * blockIdx.x + threadIdx.x;
if(offset < N)
{
c[offset] = a[offset] + b[offset];
}
}