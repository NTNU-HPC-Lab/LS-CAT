#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_MM(float *a, float *b, float *c, int wA, int wB, int hA)
{
// global index
int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // col
int gidy = blockDim.y * blockIdx.y + threadIdx.y;  // row

if(gidx < wB && gidy < hA)
{
float sum = 0.f;
for(int k=0; k<wA; k++)
{
// Multiply row of A by column of B
sum += a[gidy*wA + k] * b[k*wB +gidx];
}
c[gidy * wB + gidx] = sum;
}
}