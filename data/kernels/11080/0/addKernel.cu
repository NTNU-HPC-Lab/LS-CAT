#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This example explains how to divide the host and
* device code into separate files using vector addition
*/
#define N 64





__global__ void addKernel(float *a,float *b) {
int idx=threadIdx.x+blockIdx.x*blockDim.x;

if(idx>=N) return;
a[idx]+=b[idx];
}