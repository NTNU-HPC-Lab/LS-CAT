#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float * w_vect, float * train, float * partition, int rows, int cols){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int i=0;
float temp = 0;
for(i = 0; i<cols; i++){
temp += w_vect[i]*train[i*rows+tid];
}
partition[tid] = temp;
}