#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill(int * v, std::size_t size)
{
auto id = blockIdx.x * blockDim.x + threadIdx.x;

if( id < size)
{
v [ id ] = id;
}

}