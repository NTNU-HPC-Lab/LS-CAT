#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill(int * m, std::size_t w ,  std::size_t h)
{
auto idx = blockIdx.x * blockDim.x + threadIdx.x;
auto idy = blockIdx.y * blockDim.y + threadIdx.y;


if( idx < w && idy <h )
{
m [ idy * w + idx ] = idy * w + idx;
}

}