#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMat( float * mA_d, float * mB_d, std::size_t w, std::size_t h )
{
auto x = blockDim.x * blockIdx.x + threadIdx.x;
auto y = blockDim.y * blockIdx.y + threadIdx.y;

if( x < w && y < h )
{
mA_d[ y * w + x ] += mB_d[ y * w + x ];
}
}