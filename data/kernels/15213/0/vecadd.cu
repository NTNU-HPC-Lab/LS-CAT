#include "hip/hip_runtime.h"
#include "includes.h"




__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
auto tid = threadIdx.x;
v0[ tid ] += v1[ tid ];
}