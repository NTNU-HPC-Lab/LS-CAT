#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill( int * v, std::size_t size )
{
// Get the id of the thread ( 0 -> 99 ).
auto tid = threadIdx.x;
// Each thread fills a single element of the array.
v[ tid ] = tid;
}