#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd( int * m0, int * m1, std::size_t w, std::size_t h )
{
auto i = blockIdx.x * blockDim.x + threadIdx.x;
auto j = blockIdx.y * blockDim.y + threadIdx.y;
if( i < w && j < h )
m0[ i * w + j ] +=  m1[ i * w + j ];// i * w + j;
}