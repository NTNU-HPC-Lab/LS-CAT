#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
auto tid = blockIdx.x * blockDim.x + threadIdx.x;
if( tid < size )
{
v0[ tid ] += v1[ tid ];
}
}