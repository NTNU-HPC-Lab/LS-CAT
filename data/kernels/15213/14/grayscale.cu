#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
auto i = blockIdx.x * blockDim.x + threadIdx.x;
auto j = blockIdx.y * blockDim.y + threadIdx.y;
if( i < cols && j < rows ) {
g[ j * cols + i ] = (
307 * rgb[ 3 * ( j * cols + i ) ]
+ 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
+ 113 * rgb[  3 * ( j * cols + i ) + 2 ]
) / 1024;
}
}