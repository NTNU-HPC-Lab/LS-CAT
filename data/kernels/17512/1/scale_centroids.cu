#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void scale_centroids(int d, int k, int* counts, double* centroids) {
int global_id_x = threadIdx.x + blockIdx.x * blockDim.x;
int global_id_y = threadIdx.y + blockIdx.y * blockDim.y;
if ((global_id_x < d) && (global_id_y < k)) {
int count = counts[global_id_y];
//To avoid introducing divide by zero errors
//If a centroid has no weight, we'll do no normalization
//This will keep its coordinates defined.
if (count < 1) {
count = 1;
}
double scale = 1.0/double(count);
centroids[global_id_x + d * global_id_y] *= scale;
}
}