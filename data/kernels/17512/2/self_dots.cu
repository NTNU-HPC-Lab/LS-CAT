#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void self_dots(int n, int d, double* data, double* dots) {
double accumulator = 0;
int global_id = blockDim.x * blockIdx.x + threadIdx.x;

if (global_id < n) {
for (int i = 0; i < d; i++) {
double value = data[i + global_id * d];
accumulator += value * value;
}
dots[global_id] = accumulator;
}
}