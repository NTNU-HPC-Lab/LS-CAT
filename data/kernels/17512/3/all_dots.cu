#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void all_dots(int n, int k, double* data_dots, double* centroid_dots, double* dots) {
__shared__ double local_data_dots[32];
__shared__ double local_centroid_dots[32];

int data_index = threadIdx.x + blockIdx.x * blockDim.x;
if ((data_index < n) && (threadIdx.y == 0)) {
local_data_dots[threadIdx.x] = data_dots[data_index];
}




int centroid_index = threadIdx.x + blockIdx.y * blockDim.y;
if ((centroid_index < k) && (threadIdx.y == 1)) {
local_centroid_dots[threadIdx.x] = centroid_dots[centroid_index];
}

__syncthreads();

centroid_index = threadIdx.y + blockIdx.y * blockDim.y;
if ((data_index < n) && (centroid_index < k)) {
dots[data_index + centroid_index * n] = local_data_dots[threadIdx.x] +
local_centroid_dots[threadIdx.y];
}
}