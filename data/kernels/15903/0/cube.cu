#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void cube(float* d_out, float* d_in) {
int idx = threadIdx.x;
float f = d_in[idx];
d_out[idx] = f * f * f;
}