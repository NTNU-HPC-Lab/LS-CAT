#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(float *c, float *a, float *b, int size)
{
int i = blockIdx.x * blockDim.x *blockDim.y  + blockDim.x * threadIdx.y * threadIdx.x;
while(i < size)
{
c[i] = a[i] + b[i];
i += gridDim.x * blockDim.x * blockDim.y;
}

}