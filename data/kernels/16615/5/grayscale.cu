#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscale(unsigned char * data_rgb, unsigned char * data_gray, std::size_t rows, std::size_t cols)
{
auto i = blockIdx.x * blockDim.x + threadIdx.x;
auto j = blockIdx.y * blockDim.y + threadIdx.y;

if( i < cols && j < rows )
{
data_gray[ j * cols + i ] = (
307 * data_rgb[ 3 * (j * cols + i) ]
+   604 * data_rgb[ 3 * (j * cols + i) + 1 ]
+   113 * data_rgb[ 3 * (j * cols + i) + 2 ]
) / 1024;
}
}