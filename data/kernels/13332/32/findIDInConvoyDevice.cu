#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findIDInConvoyDevice(Convoy* d_convoy, int* d_IDIncluded, int id1, int id2)
{
//check whether thread is in bounds
if(((threadIdx.x < d_convoy[blockIdx.x].endIndexID)  && (d_convoy[blockIdx.x].endIndexID > d_convoy[blockIdx.x].startIndexID)) || ((d_convoy[blockIdx.x].endIndexID < d_convoy[blockIdx.x].startIndexID) && (threadIdx.x != d_convoy[blockIdx.x].endIndexID)))
{
int index = blockIdx.x*2;
//init memory
d_IDIncluded[index] = INT_MAX;
d_IDIncluded[index+1] = INT_MAX;
__syncthreads();
//check and write results
int result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == id1);
if(result)
{
atomicMin(&(d_IDIncluded[index]), threadIdx.x);
}
result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == id2);
if(result)
{
atomicMin(&(d_IDIncluded[index+1]), threadIdx.x);
}
//if current convoy is the ego convoy, mark it with INT_MIN
result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == -1);
if(result)
{
atomicMin(&(d_IDIncluded[index+1]), INT_MIN);
atomicMin(&(d_IDIncluded[index]), INT_MIN);
}
}
}