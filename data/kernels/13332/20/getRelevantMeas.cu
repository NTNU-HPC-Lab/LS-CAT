#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getRelevantMeas(cartesian_segment* carSegs, laserdata_cartesian* d_laser, unsigned long long* dist)
{
int index = blockIdx.x*3;
//first and last entry is trivial
d_laser[index] = carSegs[blockIdx.x].measures[0];
d_laser[index+2] = carSegs[blockIdx.x].measures[carSegs[blockIdx.x].numberOfMeasures-1];
unsigned long long tmp;
//check whether thread is out of bounds
if(threadIdx.x < carSegs[blockIdx.x].numberOfMeasures)
{
//compute distance for current position
float x = carSegs[blockIdx.x].measures[threadIdx.x].x;
float y = carSegs[blockIdx.x].measures[threadIdx.x].y;
tmp = sqrtf(x*x + y*y)*10000;
//write to shared memory
atomicMin(&(dist[blockIdx.x]), tmp);
__syncthreads();
if(dist[blockIdx.x] == tmp)
{
//own position is neareast -> write to out array
d_laser[index+1] = carSegs[blockIdx.x].measures[threadIdx.x];
}
}
}