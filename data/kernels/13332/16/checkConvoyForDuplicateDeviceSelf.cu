#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkConvoyForDuplicateDeviceSelf(Convoy* d_convoy, bool* d_duplicate)
{
if(((threadIdx.x < d_convoy[blockIdx.x].endIndexTracks)  && (d_convoy[blockIdx.x].endIndexTracks > d_convoy[blockIdx.x].startIndexTracks)) || ((d_convoy[blockIdx.x].endIndexTracks < d_convoy[blockIdx.x].startIndexTracks) && (threadIdx.x != d_convoy[blockIdx.x].endIndexTracks)))
{
d_duplicate[blockIdx.x] = true;
bool result = (d_convoy[blockIdx.x].tracks[threadIdx.x].x != 0.5f);
if(!result)
{
d_duplicate[blockIdx.x] = d_duplicate[blockIdx.x] && result;
}
}
}