#include "hip/hip_runtime.h"
#include "PointCellDevice.cuh"

/*
 * data* = [stateVector | stateCopy | F | P | H | R | K | I | Q | S | tmp | tmp2]
 */
PointCellDevice::PointCellDevice()
{
	initializeMemory();
	subInvtl = 0.5;
}

PointCellDevice::~PointCellDevice()
{
}
/*
 * set initial values to all matrices and vectors
 */
__host__ __device__ void PointCellDevice::initializeMemory()
{
	//initialize data to 0
	for(int i=0; i<260; i++)
	{
		data[i] = 0;
	}

	for(int i=0; i<5; i++)
	{
		//P
		data[35 + i*5 + i] = 1000;
		//F
		data[10 + i*5 + i] = 1;
		//I
		data[135 + i*5 + i] = 1;
		//H
		data[60 + i*5 + i] = 1;
	}
	//F(2,4)
	data[10 + 2*5 +4] = TIMESTAMP;

	//Q
	data[160] = 0.000006f;
	data[160 + 1*5 + 1] = 0.000006f;
	data[160 + 2*5 + 2] = 0.0004f;
	data[160 + 3*5 + 3] = 0.03097f;
	data[160 + 4*5 + 4] = 0.0004f;

	//R
	data[85] = 0.36f;
	data[85 + 1*5 + 1] = 0.36f;
	data[85 + 2*5 + 2] = 0.5f;
	data[85 + 3*5 + 3] = 0.1f;
	data[85 + 4*5 + 4] = 0.1f;
}
/*
 * perfrom kalman filter predict step
 */
__host__ __device__ void PointCellDevice::predict()
{
	//store copy of stateVector
	for(int i=0; i<5; i++)
	{
		data[i+5] = data[i];
	}
	//estimate new state
	computeF();
	//compute new state covariance
	computeCovarianceF();

	float tmp = 0;
	// Tmp = F*P
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getF(i,k)*getP(k,j);
			}
			writeTmp(i,j, tmp);
		}
	}

	//P = Tmp*F_t
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getTmp(i,k)*getF(j,k);
			}
			writeP(i,j, tmp);
		}
	}

	//P = P+Q
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = getP(i,j) + getQ(i,j);
			writeP(i,j, tmp);
		}
	}
}
/*
 * estimates new state
 */
__host__ __device__ void PointCellDevice::computeF()
{
	float x = getX();
	float y = getY();
	float theta = getTheta();
	float velocity = getVelocity();
	float phi = getPhi();

	float predictedX, predictedY, predictedTheta, predictedVel,predictedPhi;

	if(phi > 0.0001)
	{
		predictedX = (velocity/phi) * (sinf(phi*TIMESTAMP + theta) - sinf(theta)) + x;
		predictedY = (velocity/phi) * (-cosf(phi*TIMESTAMP + theta) + cosf(theta)) + y;
		predictedTheta = phi*TIMESTAMP + theta;
		predictedVel = velocity;
		predictedPhi = phi;
	}
	else
	{
		predictedX = x + velocity * TIMESTAMP * cosf(theta);
		predictedY = y + velocity * TIMESTAMP * sinf(theta);
		predictedTheta = theta;
		predictedVel = velocity;
		predictedPhi = 0.00001;
	}

	setX(predictedX);
	setY(predictedY);
	setTheta(predictedTheta);
	setVelocity(predictedVel);
	setPhi(predictedPhi);
}
/*
 * computes new state covariance
 */
__host__ __device__ void PointCellDevice::computeCovarianceF()
{
	float theta = getTheta();
	float velocity = getVelocity();
	float phi = getPhi();

	float f12, f13, f14, f22, f23, f24;

	f12 = (velocity/phi) * (-cosf(theta) + cosf(TIMESTAMP*phi + theta));
	f13 = (1/phi) * (sinf(phi*TIMESTAMP + theta) - sinf(theta));
	f14 = (((TIMESTAMP*velocity)/phi) * cosf(TIMESTAMP*phi + theta)) - ((velocity/(phi*phi)) * (sinf(phi*TIMESTAMP + theta) - sinf(theta)));

	f22 = (velocity/phi) * (sinf(phi*TIMESTAMP + theta) - sinf(theta));
	f23 = (1/phi) * (-cosf(phi*TIMESTAMP + theta) + cosf(theta));
	f24 = (((TIMESTAMP*velocity)/phi) * sinf(TIMESTAMP*phi + theta)) - ((velocity/(phi*phi)) * (-cosf(phi*TIMESTAMP + theta) + cosf(theta)));

	writeF(0,2,f12);
	writeF(0,3,f13);
	writeF(0,4,f14);
	writeF(1,2,f22);
	writeF(1,3,f23);
	writeF(1,4,f24);
}
/*
 * perfomrs kalman filter update step with given new state
 */
__host__ __device__ void PointCellDevice::update(float* newState)
{
	float velocity, phi;
	float xNew = newState[0];
	float yNew = newState[1];
	float thetaNew = newState[2];

	float x = data[5];
	float y = data[6];
	float theta = data[7];
	//first compute yawrate and velocity based in new and old position
	velocity = sqrtf((xNew - x) * (xNew - x) + (yNew - y)*(yNew - y)) / TIMESTAMP;
	phi = (thetaNew-theta) / TIMESTAMP;

	setVelocity(velocity);
	setPhi(phi);
	float tmp = 0;

	//tmp = H*P
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getH(i,k)*getP(k,j);
			}
			writeTmp(i,j, tmp);
		}
	}

	//S = tmp*H_t
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getTmp(i,k)*getH(j,k);
			}
			writeS(i,j, tmp);
		}
	}

	//S = S+R
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = getS(i,j) + getR(i,j);
			writeS(i,j, tmp);
		}
	}

	//tmp = P*H_t
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getP(i,k)*getH(j,k);
			}
			writeTmp(i,j, tmp);
		}
	}

	invertS();

	//K = tmp*S_i
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getTmp(i,k)*getS(k,j);
			}
			writeK(i,j, tmp);
		}
	}

	//tmp = K*(newState-stateVector)
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<1; j++)
		{
			tmp = 0;
			tmp += getK(i,0)*(xNew-getX());
			tmp += getK(i,1)*(yNew-getY());
			tmp += getK(i,2)*(thetaNew-getTheta());
			tmp += getK(i,3)*(velocity-getVelocity());
			tmp += getK(i,4)*(phi-getPhi());
			writeTmp(i,j, tmp);
		}
	}

	//stateVector = stateVector + tmp
	setX(getX() + getTmp(0,0));
	setY(getY() + getTmp(1,0));
	setTheta(getTheta() + getTmp(2,0));
	setVelocity(getVelocity() + getTmp(3,0));
	setPhi(getPhi() + getTmp(4,0));

	//tmp = K*H
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getK(i,k)*getH(k,j);
			}
			writeTmp(i,j, tmp);
		}
	}

	//tmp = I - tmp
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = getI(i,j) - getTmp(i,j);
			writeTmp(i,j, tmp);
		}
	}

	//tmp2 = tmp*P
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			tmp = 0;
			for(int k=0; k<5; k++)
			{
				tmp += getTmp(i,k)*getP(k,j);
			}
			writeTmp2(i,j, tmp);
		}
	}

	for(int i=0; i<5;i++)
	{
		for(int j=0; j<5; j++)
		{
			writeP(i,j, getTmp2(i,j));
		}
	}

}
/*
 * performs the inversion of S
 */
__host__ __device__ void PointCellDevice::invertS()
{
	//Concatenate IdentityMatrix to the right of S
	float toInvert[50];
	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			toInvert[i*10 +j] = getS(i,j);
		}
	}
	for(int i=0; i<5; i++)
	{
		for(int j=5; j<10; j++)
		{
			if(j-i == 5)
			{
				toInvert[i*10 +j] = 1;
			}
			else
			{
				toInvert[i*10 +j] = 0;
			}
		}
	}
	reducedRowEcholon(toInvert);

}

__host__ __device__ void PointCellDevice::reducedRowEcholon(float* toInvert)
{
    float const ZERO = static_cast<float>( 0 );
    int order[5];
    int rows = 5;
    int columns = 10;
    for(int i=0; i<5; i++)
    {
    	order[i] = i;
    }
    // For each row...
    for ( unsigned rowIndex = 0; rowIndex < rows; ++rowIndex )
    {
      // Reorder the rows.
      reorder(toInvert, order);

      unsigned row = order[ rowIndex ];

      // Divide row down so first term is 1.
      unsigned column = getLeadingZeros( row , toInvert);
      float divisor = toInvert[(row * columns) + column];
      if ( ZERO != divisor )
      {
        divideRow(toInvert, row, divisor );

        // Subtract this row from all subsequent rows.
        for ( unsigned subRowIndex = ( rowIndex + 1 ); subRowIndex < rows; ++subRowIndex )
        {
          unsigned subRow = order[ subRowIndex ];
          if ( ZERO != toInvert[(subRow * columns) + column] )
            rowOperation
            (
              toInvert,
              subRow,
              row,
              -toInvert[(subRow * columns) + column]
            );
        }
      }

    }

    // Back substitute all lower rows.
    for ( unsigned rowIndex = ( rows - 1 ); rowIndex > 0; --rowIndex )
    {
      unsigned row = order[ rowIndex ];
      unsigned column = getLeadingZeros( row ,toInvert);
      for ( unsigned subRowIndex = 0; subRowIndex < rowIndex; ++subRowIndex )
      {
        unsigned subRow = order[ subRowIndex ];
        rowOperation
        (
          toInvert,
          subRow,
          row,
          -toInvert[(subRow * columns) + column]
        );
      }
    }
    getSubMatrix(toInvert,0, 4, 5, 9, order);
}
__host__ __device__ void PointCellDevice::reorder(float* toInvert, int* order)
{
    unsigned zeros[5];
    int rows = 5;
    for ( unsigned row = 0; row < rows; ++row )
    {
      order[ row ] = row;
      zeros[ row ] = getLeadingZeros(row, toInvert);
    }

    for ( unsigned row = 0; row < (rows-1); ++row )
    {
      unsigned swapRow = row;
      for ( unsigned subRow = row + 1; subRow < rows; ++subRow )
      {
        if ( zeros[ order[ subRow ] ] < zeros[ order[ swapRow ] ] )
          swapRow = subRow;
      }

      unsigned hold    = order[ row ];
      order[ row ]     = order[ swapRow ];
      order[ swapRow ] = hold;
    }
}
__host__ __device__ void PointCellDevice::divideRow(float* toInvert, int row, float divisor)
{
    for ( unsigned column = 0; column < 10; ++column )
    {
      toInvert[ (row * 10) + column] /= divisor;
    }
}
__host__ __device__ void PointCellDevice::rowOperation(float* toInvert, int row, int addRow, float scale)
{
	int columns = 10;
    for ( unsigned column = 0; column < columns; ++column )
    {
      toInvert[ (row * columns) + column] += toInvert[ (addRow * columns) + column] * scale;
    }
}

__host__ __device__ unsigned PointCellDevice::getLeadingZeros(unsigned row, float* toInvert) const
{
	  float const ZERO = static_cast< float >( 0 );
	  unsigned column = 0;
	  while ( ZERO == toInvert[ (row * 10) + column] )
	  {
	    ++column;
	  }
	  return column;
}

__host__ __device__ void PointCellDevice::getSubMatrix(float* toInvert, unsigned startRow,unsigned endRow,unsigned startColumn,unsigned endColumn, int* newOrder)
{
	int columns = 10;
    for ( unsigned row = startRow; row <= endRow; ++row )
    {
      unsigned subRow;
      if ( NULL == newOrder )
        subRow = row;
      else
        subRow = newOrder[ row ];

      for ( unsigned column = startColumn; column <= endColumn; ++column )
      {
    	 writeS((row - startRow),(column - startColumn), toInvert[ (subRow * columns) + column]);
      }
    }

}
__host__ __device__ int PointCellDevice::getID()
{
	return ID;
}
__host__ __device__ void PointCellDevice::setID(int id)
{
	ID = id;
}
__host__ __device__ float PointCellDevice::getX()
{
	return data[0];
}
__host__ __device__ float PointCellDevice::getY()
{
	return data[1];
}
__host__ __device__ float PointCellDevice::getTheta()
{
	return data[2];
}
__host__ __device__ float PointCellDevice::getVelocity()
{
	return data[3];
}
__host__ __device__ float PointCellDevice::getPhi()
{
	return data[4];
}

__host__ __device__ void PointCellDevice::setX(float x)
{
	data[0] = x;
}
__host__ __device__ void PointCellDevice::setY(float y)
{
	data[1] = y;
}
__host__ __device__ void PointCellDevice::setTheta(float theta)
{
	data[2] = theta;
}
__host__ __device__ void PointCellDevice::setVelocity(float velocity)
{
	data[3] = velocity;
}
__host__ __device__ void PointCellDevice::setPhi(float phi)
{
	data[4] = phi;
}

__host__ __device__ void PointCellDevice::writeP(int row, int col, float value)
{
	data[35 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeF(int row, int col, float value)
{
	data[10 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeH(int row, int col, float value)
{
	data[60 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeR(int row, int col, float value)
{
	data[85 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeK(int row, int col, float value)
{
	data[110 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeI(int row, int col, float value)
{
	data[135 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeQ(int row, int col, float value)
{
	data[160 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeS(int row, int col, float value)
{
	data[185 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeTmp(int row, int col, float value)
{
	data[210 + row*5 + col] = value;
}

__host__ __device__ void PointCellDevice::writeTmp2(int row, int col, float value)
{
	data[235 + row*5 + col] = value;
}

__host__ __device__ float PointCellDevice::getP(int row, int col)
{
	return data[35 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getF(int row, int col)
{
	return data[10 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getH(int row, int col)
{
	return data[60 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getR(int row, int col)
{
	return data[85 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getK(int row, int col)
{
	return data[110 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getI(int row, int col)
{
	return data[135 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getQ(int row, int col)
{
	return data[160 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getS(int row, int col)
{
	return data[185 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getTmp(int row, int col)
{
	return data[210 + row*5 + col];
}

__host__ __device__ float PointCellDevice::getTmp2(int row, int col)
{
	return data[235 + row*5 + col];
}
