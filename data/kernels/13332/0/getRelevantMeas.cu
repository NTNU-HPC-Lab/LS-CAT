#include "hip/hip_runtime.h"
#include "includes.h"
/*
* DataReader.cpp
*
*  Created on: 06.06.2016
*      Author: Sebastian Reinhart
*/



__global__ void getRelevantMeas(cartesian_segment* carSegs, laserdata_cartesian* d_laser, unsigned long long* dist)
{
int index = blockIdx.x*3;
d_laser[index] = carSegs[blockIdx.x].measures[0];
d_laser[index+2] = carSegs[blockIdx.x].measures[carSegs[blockIdx.x].numberOfMeasures-1];
unsigned long long tmp;
if(threadIdx.x < carSegs[blockIdx.x].numberOfMeasures)
{
float x = carSegs[blockIdx.x].measures[threadIdx.x].x;
float y = carSegs[blockIdx.x].measures[threadIdx.x].y;
tmp = sqrtf(x*x + y*y)*10000;
atomicMin(&(dist[blockIdx.x]), tmp);
__syncthreads();
if(dist[blockIdx.x] == tmp)
{
d_laser[index+1] = carSegs[blockIdx.x].measures[threadIdx.x];
}
}
}