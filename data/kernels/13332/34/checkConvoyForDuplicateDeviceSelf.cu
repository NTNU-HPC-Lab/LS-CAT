#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkConvoyForDuplicateDeviceSelf(Convoy* d_convoy, bool* d_duplicate)
{
//check whether thread is in bounds
if(((threadIdx.x < d_convoy[blockIdx.x].endIndexTracks)  && (d_convoy[blockIdx.x].endIndexTracks > d_convoy[blockIdx.x].startIndexTracks)) || ((d_convoy[blockIdx.x].endIndexTracks < d_convoy[blockIdx.x].startIndexTracks) && (threadIdx.x != d_convoy[blockIdx.x].endIndexTracks)))
{
d_duplicate[blockIdx.x] = true;
bool result = (d_convoy[blockIdx.x].tracks[threadIdx.x].x != 0.5);
if(!result)
{
d_duplicate[blockIdx.x] = d_duplicate[blockIdx.x] && result;
}
}
}