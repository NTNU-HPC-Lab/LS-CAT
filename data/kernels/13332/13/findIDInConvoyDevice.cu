#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findIDInConvoyDevice(Convoy* d_convoy, int* d_IDIncluded, int id1, int id2)
{
if(((threadIdx.x < d_convoy[blockIdx.x].endIndexID)  && (d_convoy[blockIdx.x].endIndexID > d_convoy[blockIdx.x].startIndexID)) || ((d_convoy[blockIdx.x].endIndexID < d_convoy[blockIdx.x].startIndexID) && (threadIdx.x != d_convoy[blockIdx.x].endIndexID)))
{
int index = blockIdx.x*2;
d_IDIncluded[index] = INT_MAX;
d_IDIncluded[index+1] = INT_MAX;
__syncthreads();
int result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == id1);
if(result)
{
atomicMin(&(d_IDIncluded[index]), threadIdx.x);
}
result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == id2);
if(result)
{
atomicMin(&(d_IDIncluded[index+1]), threadIdx.x);
}
result = (d_convoy[blockIdx.x].participatingVehicles[threadIdx.x] == -1);
if(result)
{
atomicMin(&(d_IDIncluded[index+1]), INT_MIN);
atomicMin(&(d_IDIncluded[index]), INT_MIN);
}

}
}