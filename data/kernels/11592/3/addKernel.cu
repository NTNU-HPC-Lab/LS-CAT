#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a)
{
int i = threadIdx.x;
extern __shared__ int smem[];
smem[i] = a[i];
__syncthreads();


if(i == 0)  // 0号线程做平方和
{
c[0] = 0;
for(int d = 0; d < 5; d++)
{
c[0] += smem[d] * smem[d];

}
}
if(i == 1)//1号线程做累加
{
c[1] = 0;
for(int d = 0; d < 5; d++)
{
c[1] += smem[d];
}
}
if(i == 2)  //2号线程做累乘
{
c[2] = 1;
for(int d = 0; d < 5; d++)
{
c[2] *= smem[d];
}
}
}