#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = blockIdx.x;
c[i] = a[i] + b[i];
}