#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeLocalMemory(float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
int tid=threadIdx.x, localSize=blockDim.x, litems=2048/localSize;
int s = tid;
__shared__ float lbuf[2048];
for (j=0 ; j<repeat ; ++j)
{
lbuf[(s+0)&(2047)] = gid;
lbuf[(s+1)&(2047)] = gid;
lbuf[(s+2)&(2047)] = gid;
lbuf[(s+3)&(2047)] = gid;
lbuf[(s+4)&(2047)] = gid;
lbuf[(s+5)&(2047)] = gid;
lbuf[(s+6)&(2047)] = gid;
lbuf[(s+7)&(2047)] = gid;
lbuf[(s+8)&(2047)] = gid;
lbuf[(s+9)&(2047)] = gid;
lbuf[(s+10)&(2047)] = gid;
lbuf[(s+11)&(2047)] = gid;
lbuf[(s+12)&(2047)] = gid;
lbuf[(s+13)&(2047)] = gid;
lbuf[(s+14)&(2047)] = gid;
lbuf[(s+15)&(2047)] = gid;
s = (s+16)&(2047);
}
__syncthreads();
for (j=0 ; j<litems ; ++j)
output[gid] = lbuf[tid];
}