#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward1b(float *zs,int bound,float *weights,int w_off,float *activations) {

int i;

zs[(blockIdx.x*blockDim.x)+threadIdx.x]=0.0;
for (i=0; i<bound; i++) {
zs[(blockIdx.x*blockDim.x)+threadIdx.x]+=weights[w_off+(threadIdx.x*bound)+i]*activations[(blockIdx.x*bound)+i];
}
}