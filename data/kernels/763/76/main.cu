#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "BFS_kernel_one_block.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
volatile unsigned int *frontier = NULL;
hipMalloc(&frontier, XSIZE*YSIZE);
unsigned int frontier_len = 1;
volatile unsigned int *cost = NULL;
hipMalloc(&cost, XSIZE*YSIZE);
volatile int *visited = NULL;
hipMalloc(&visited, XSIZE*YSIZE);
unsigned int *edgeArray = NULL;
hipMalloc(&edgeArray, XSIZE*YSIZE);
unsigned int *edgeArrayAux = NULL;
hipMalloc(&edgeArrayAux, XSIZE*YSIZE);
unsigned int numVertices = 1;
unsigned int numEdges = 1;
volatile unsigned int *frontier_length = NULL;
hipMalloc(&frontier_length, XSIZE*YSIZE);
unsigned int num_p_per_mp = 1;
unsigned int w_q_size = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
BFS_kernel_one_block<<<gridBlock,threadBlock>>>(frontier,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,num_p_per_mp,w_q_size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
BFS_kernel_one_block<<<gridBlock,threadBlock>>>(frontier,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,num_p_per_mp,w_q_size);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
BFS_kernel_one_block<<<gridBlock,threadBlock>>>(frontier,frontier_len,cost,visited,edgeArray,edgeArrayAux,numVertices,numEdges,frontier_length,num_p_per_mp,w_q_size);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}