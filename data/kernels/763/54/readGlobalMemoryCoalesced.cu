#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readGlobalMemoryCoalesced(float *data, float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
float sum = 0;
int s = gid;
for (j=0 ; j<repeat ; ++j)
{
float a0 = data[(s+0)&(size-1)];
float a1 = data[(s+32768)&(size-1)];
float a2 = data[(s+65536)&(size-1)];
float a3 = data[(s+98304)&(size-1)];
float a4 = data[(s+131072)&(size-1)];
float a5 = data[(s+163840)&(size-1)];
float a6 = data[(s+196608)&(size-1)];
float a7 = data[(s+229376)&(size-1)];
float a8 = data[(s+262144)&(size-1)];
float a9 = data[(s+294912)&(size-1)];
float a10 = data[(s+327680)&(size-1)];
float a11 = data[(s+360448)&(size-1)];
float a12 = data[(s+393216)&(size-1)];
float a13 = data[(s+425984)&(size-1)];
float a14 = data[(s+458752)&(size-1)];
float a15 = data[(s+491520)&(size-1)];
sum += a0+a1+a2+a3+a4+a5+a6+a7+a8+a9+a10+a11+a12+a13+a14+a15;
s = (s+524288)&(size-1);
}
output[gid] = sum;
}