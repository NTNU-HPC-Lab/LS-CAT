#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop3a(float *delta_nabla_b,int b_off,int bound,int b_off_old,float *weights,int w_off_old) {

int j;

delta_nabla_b[b_off+threadIdx.x]=0.0;
for (j=0; j<bound; j++) {
delta_nabla_b[b_off+threadIdx.x]+=delta_nabla_b[b_off_old+j]*weights[w_off_old+(j*blockDim.x)+threadIdx.x];
}
}