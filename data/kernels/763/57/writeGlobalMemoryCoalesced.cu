#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeGlobalMemoryCoalesced(float *output, int size, int repeat)
{
int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
int s = gid;
for (j=0 ; j<repeat ; ++j)
{
output[(s+0)&(size-1)] = gid;
output[(s+32768)&(size-1)] = gid;
output[(s+65536)&(size-1)] = gid;
output[(s+98304)&(size-1)] = gid;
output[(s+131072)&(size-1)] = gid;
output[(s+163840)&(size-1)] = gid;
output[(s+196608)&(size-1)] = gid;
output[(s+229376)&(size-1)] = gid;
output[(s+262144)&(size-1)] = gid;
output[(s+294912)&(size-1)] = gid;
output[(s+327680)&(size-1)] = gid;
output[(s+360448)&(size-1)] = gid;
output[(s+393216)&(size-1)] = gid;
output[(s+425984)&(size-1)] = gid;
output[(s+458752)&(size-1)] = gid;
output[(s+491520)&(size-1)] = gid;
s = (s+524288)&(size-1);
}
}