#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
return;

long int id = (k * c_Size.y + j) * c_Size.x + i;
long int id_x = (k * c_Size.y + j) * c_Size.x + i + 1;
long int id_y = (k * c_Size.y + j + 1) * c_Size.x + i;
long int id_z = ((k + 1) * c_Size.y + j) * c_Size.x + i;

if (i != (c_Size.x - 1))
grad_x[id] -= ((in[id_x] - in[id]) / c_Spacing.x);
if (j != (c_Size.y - 1))
grad_y[id] -= ((in[id_y] - in[id]) / c_Spacing.y);
if (k != (c_Size.z - 1))
grad_z[id] -= ((in[id_z] - in[id]) / c_Spacing.z);
}