#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atemp(double* A, double* y, double* tmp, int NX, int NY)
{
int j;
int i = blockDim.x * blockIdx.x + threadIdx.x;
// Α(T)*temp
if (i <= NY){
for (j = 0; j < NX; j++) {
y[i] = y[i] + A[i + j*NY] * tmp[j];
}
}
}