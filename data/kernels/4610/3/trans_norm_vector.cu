#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void trans_norm_vector(double* A, double* x, double* y, double* tmp, int NX, int NY)
{
int j;
int i = blockDim.x * blockIdx.x + threadIdx.x;

tmp[i] = 0;
//Α*Χ
for (j = 0; j < NY; j++) {
tmp[i] = tmp[i] + A[i*NY + j] * x[j];
}

}