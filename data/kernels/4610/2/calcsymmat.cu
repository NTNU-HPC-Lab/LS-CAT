#include "hip/hip_runtime.h"
#include "includes.h"

#define FLOAT_N 3214212.01

__global__ void calcsymmat(double* d_data, double* d_symmat, int M, int N)
{
int	i, j2;
int j1 = blockDim.x * blockIdx.x + threadIdx.x+1;
if (j1<=(M+1)) {
for (j2 = j1; j2 < (M+1); j2++) {
d_symmat[j1*(M+1) + j2] = 0.0;
for (i = 1; i < N+1; i++) {
d_symmat[j1*(M+1) + j2] += d_data[i*(M+1) + j1] * d_data[i*(M+1) + j2];
}
d_symmat[j2*(M+1) + j1] = d_symmat[j1*(M+1) + j2];
}
}
}