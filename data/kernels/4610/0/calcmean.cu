#include "hip/hip_runtime.h"
#include "includes.h"

#define FLOAT_N 3214212.01

__global__ void calcmean(double* d_data, double* d_mean, int M, int N)
{
int	i;
int j = blockDim.x * blockIdx.x + threadIdx.x+1;
if (j<=(M+1)) {
d_mean[j] = 0.0;
for (i = 1; i < (N+1); i++) {
d_mean[j] += d_data[i*(M+1) + j];
}
d_mean[j] /= FLOAT_N;
}
}