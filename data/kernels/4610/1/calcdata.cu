#include "hip/hip_runtime.h"
#include "includes.h"

#define FLOAT_N 3214212.01

__global__ void calcdata(double* d_data, double* d_mean, int M, int N)
{
int j;
int i = blockDim.x * blockIdx.x + threadIdx.x+1;
if (i<=(N+1)) {
for (j = 1; j < (M+1); j++) {
d_data[i*(M+1) + j] -= d_mean[j];
}
}
}