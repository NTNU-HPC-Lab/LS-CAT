#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Convolution(double* A, double* B, int I, int J)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
double c11, c12, c13, c21, c22, c23, c31, c32, c33;

c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
c13 = +0.4;  c23 = +0.7;  c33 = +0.1;

if (i>J && i<I*J-J && (i%J!=0) && ((i+1)%J!=0)) {
B[i] = c11 * A[i-J-1]  +  c12 * A[i-1]  +  c13 * A[i+J-1]
+ c21 * A[i-J]  +  c22 * A[i]  +  c23 * A[i+J]
+ c31 * A[i-J+1]  +  c32 * A[i+1]  +  c33 * A[i+J+1];
}

}