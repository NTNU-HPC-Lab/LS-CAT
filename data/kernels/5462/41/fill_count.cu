#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_count(float * prp_0,int sz)
{
// Thread index
int tx = threadIdx.x + blockIdx.x * blockDim.x;
int ty = threadIdx.y + blockIdx.y * blockDim.y;
int tz = threadIdx.z + blockIdx.z * blockDim.z;

prp_0[tz*sz*sz + ty*sz + tx] = tz*sz*sz + ty*sz + tx;
}