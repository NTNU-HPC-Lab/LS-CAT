#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiplication(int *matrix_1, int *matrix_2, int *matrix_r, int m, int n, int p){

int row = threadIdx.y + blockIdx.y * blockDim.y; 	// Multiply this row...
int col = threadIdx.x + blockIdx.x * blockDim.x;	// with this column.

// Matrix multiplication as follows:
// (m x n) x (n x p) = (m x p)

int id = row * p + col;	// Index of the result matrix in which we will write.
int sum = 0;

if (row < m && col < p) {
for(int i = 0; i < n; i++) {
// In matrix_1 we keep the row and advance in the columns.
// In matrix_2 we keep the column and advance in the rows.
sum = sum + matrix_1[row * n + i] * matrix_2[i * p + col];
// row * n stays in the same row and  "+ i" advances 1 column each cicle.
// i * p advances one row each cicle and  "+ col" keeps the same col.
}
matrix_r[id] = sum;
}
}