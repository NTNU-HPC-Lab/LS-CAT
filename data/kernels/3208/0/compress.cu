#include "hip/hip_runtime.h"
#include "includes.h"
/*
*
*    Carlos Roman Rivera - A01700820
*
*    Programming Languages - Cuda Quiz
*
*/


#define N 9
#define K N/3
#define ThreadsPerBlock K
#define NumBlocks K


__global__ void compress(float *mat, int n, float *comp, int k){
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;

if (row < k && col < k) {
comp[col + row * k] = 0;
for (int i_row = 0 ; i_row < k ; i_row++) {
for (int j_col = 0 ; j_col < k ; j_col++) {
comp[col + row * k] += mat[(col + j_col) + (row + i_row) * n];
}
}
}

}