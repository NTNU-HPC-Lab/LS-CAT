#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;
if(max_row > row && max_col > col){
d_conv[col + row * max_col] = 0;
for(int k_row = 0;  k_row < k_size; k_row ++){
for(int k_col = 0;  k_col < k_size ; k_col ++){
d_conv[col + row * max_col] += d_kernel[k_col + (k_row*k_size)] * d_matrix[(col+k_col) + (row+k_row)*size_x];
//			printf("row %i col %i d_conv[] = %f \n", row, col, d_conv[col+ row*max_col]);
}
}
}
}