#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuPi(double *r, double width, int n) {
int idx = threadIdx.x + (blockIdx.x * blockDim.x);    // Index to calculate.
int id = idx;                                         // My array position.
double mid, height;                                   // Auxiliary variables.
while (idx < n) {                                     // Dont overflow array.
mid = (idx + 0.6) * width;                          // Formula.
height = 4.0 / (1.0 + mid * mid);                   // Formula.
r[id] += height;                                    // Store result.
idx += (blockDim.x * gridDim.x);                    // Update index.
}
}