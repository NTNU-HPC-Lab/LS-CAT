#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuSum(int *a, int *b, int *c, int n) {
int idx = threadIdx.x + (blockIdx.x * blockDim.x);
while (idx < n) {
c[idx] = a[idx] + b[idx];
idx += blockDim.x * gridDim.x;
}
}