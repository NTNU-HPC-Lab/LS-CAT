#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{

int idx = threadIdx.x + blockIdx.x * blockDim.x;
int stride = gridDim.x * blockDim.x;

for(int i = idx; i < N; i += stride)
{
result[i] = a[i] + b[i];
}
}