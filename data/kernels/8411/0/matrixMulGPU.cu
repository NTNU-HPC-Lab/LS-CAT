#include "hip/hip_runtime.h"
#include "includes.h"

#define N  64


/*
* This CPU function already works, and will run to create a solution matrix
* against which to verify your work building out the matrixMulGPU kernel.
*/

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
/*
* Build out this kernel.
*/
int val = 0;
int row = threadIdx.x + blockIdx.x * blockDim.x;
int col = threadIdx.y + blockIdx.y * blockDim.y;

if (row < N && col < N)
{
for ( int k = 0; k < N; ++k )
val += a[row * N + k] * b[k * N + col];
c[row * N + col] = val;
}
}