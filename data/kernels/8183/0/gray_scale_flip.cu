#include "hip/hip_runtime.h"
#include "includes.h"
/******************************
*STUDENT NAME: DAVID PARKS    *
*PROJECT: 6 - GREY SCALE FLIP *
*DUE DATE: THURS 18/10/18     *
*******************************/

#define PPM_MAGIC_1 'P'
#define PPM_MAGIC_2 '6'
#define BLOCK_SIZE 16;

struct PPM_header {
int width;
int height;
int max_color;
};
struct RGB_8 {
uint8_t r;
uint8_t g;
uint8_t b;
};//__attribute__((packed));

__global__ void gray_scale_flip(RGB_8* img, int height, int width)
{
int row = blockDim.y * blockIdx.y + threadIdx.y;
int col = blockDim.x * blockIdx.x + threadIdx.x;

if (row < height && col < width / 2)
{
int i = row * width + col;
//temp var for slip pixel
RGB_8 temp = img[(row + 1) * width - col - 1];

//computing gray value
float gray_value = 0.21 * img[i].r + 0.72 * img[i].g + 0.07 * img[i].b;
img[i].r = gray_value;
img[i].g = gray_value;
img[i].b = gray_value;

//set flip pixel to grayed current pixel
img[(row + 1) * width - col - 1] = img[i];

//set current pixel to temp pixel
img[i] = temp;

//computing gray value
gray_value = 0.21 * img[i].r + 0.72 * img[i].g + 0.07 * img[i].b;
img[i].r = gray_value;
img[i].g = gray_value;
img[i].b = gray_value;
}
}