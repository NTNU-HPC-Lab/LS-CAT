#include "hip/hip_runtime.h"
#include "includes.h"


#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0




__global__ void pointToThreadMove(int pointsCounter, int threadsInsideBlock, double dt, double *pointsInGpu, double *speedArrayInGpu)
{
/**
This function moves the thread with the right velocity readed from the file.
This function puts every point in ONE thread.
**/
int blockDimLeft=pointsCounter % blockDim.x;
if (blockIdx.x != gridDim.x - 1 || blockDimLeft > threadIdx.x)
{
int indexInArray=0;
while(indexInArray < DIMENSIONS)
{
int currentBlock=blockIdx.x * DIMENSIONS * threadsInsideBlock;
int currentThread=threadIdx.x* DIMENSIONS;
int currentGpuPoint = currentBlock + currentThread + indexInArray;
pointsInGpu[currentGpuPoint] += speedArrayInGpu[currentGpuPoint] * dt;
indexInArray++;
}
}
}