#include "hip/hip_runtime.h"
#include "includes.h"


#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0




__global__ void distanceArrCalc(int pointsCounter, int threads, double *distanceFromPointToCluster, double *pointsInGpu, double *clustersInGpu)
{
/**
This Function computes distances. Every index is a point. Every value inside an index is a distance.
**/
double distanceX = 0;
double distanceY= 0;

int threadsLeft=pointsCounter % blockDim.x;
if ((threadsLeft > threadIdx.x) || (blockIdx.x+1 != gridDim.x))
{
int offsetPointIndex=(blockIdx.x * threads + threadIdx.x)*DIMENSIONS;
int offsetClusterIndexForPoint=threadIdx.y * DIMENSIONS;

//calc X
double a=pointsInGpu[offsetPointIndex];
double b= clustersInGpu[offsetClusterIndexForPoint];
distanceX = (a - b);
distanceX*=distanceX;

//calc Y
a=pointsInGpu[offsetPointIndex+1];
b= clustersInGpu[offsetClusterIndexForPoint+1];
distanceY =(a - b);
distanceY*=distanceY;


double totalDistance=sqrt(distanceY+distanceX);
int currentPointIndexY = pointsCounter*threadIdx.y;
int currentPointIndexX=(blockIdx.x * threads + threadIdx.x);
int pointIndex=currentPointIndexY+currentPointIndexX;

distanceFromPointToCluster[pointIndex] = totalDistance;
}
}