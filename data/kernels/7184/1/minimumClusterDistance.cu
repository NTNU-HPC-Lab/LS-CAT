#include "hip/hip_runtime.h"
#include "includes.h"


#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0




__global__ void minimumClusterDistance(int threads, double *pointToClusterDistance, int *minimumPointToCluster, int pointsCounter, int clusterCounter)
{
/**
This function puts the point in the right cluster after computing smallest distances.
**/

int leftThreads=pointsCounter % blockDim.x;

if ((blockIdx.x +1 != gridDim.x) || (leftThreads > threadIdx.x))
{
int index=0;
double smallestIndex; //minimum index
double min; //minimum distance
double temp; //temp distance
int pointIndex=threads * blockIdx.x + threadIdx.x;
min = pointToClusterDistance[pointIndex];
int currentIndex;

while(index<clusterCounter)
{

currentIndex=index*pointsCounter;
temp = pointToClusterDistance[pointIndex + currentIndex];
if(temp < min)
{
smallestIndex = index;
min = temp;
}
index++;
}

minimumPointToCluster[pointIndex] = smallestIndex;
}
}