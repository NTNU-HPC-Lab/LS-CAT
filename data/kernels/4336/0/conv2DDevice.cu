#include "hip/hip_runtime.h"
#include "includes.h"
/**
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/***********************************************************************************\
* Standard Includes
\***********************************************************************************/
/* For the CUDA runtime routines (prefixed with "cuda_") */

/***********************************************************************************\
* # Defines
\***********************************************************************************/
#define TRUE (1)
#define FALSE (0)

/* Number of rows and columns of the global memory */
#define NUM_OF_GLOBAL_ROWS (16000)
#define NUM_OF_GLOBAL_COLS (16000)

/* Number of threads in each block */
#define BLOCK_SIZE (32)

/* Convolution Kernel size */
#define KERNEL_SIZE (2)

/* Number of row and columns of the local memory */
#define NUM_OF_LOCAL_ROWS (BLOCK_SIZE + (2 * KERNEL_SIZE))
#define NUM_OF_LOCAL_COLS (BLOCK_SIZE + (2 * KERNEL_SIZE))

/***********************************************************************************\
* Enums
\***********************************************************************************/
typedef enum Status_Tag
{
PASSED,
FAILED
}Status_T;

typedef enum Cuda_Event_Tag
{
ALLOCATE_DEVICE_MATRIX_A,
ALLOCATE_DEVICE_MATRIX_B,
COPY_MATRIX_A_FROM_HOST_TO_DEVICE,
LAUNCH_KERNEL_CONV2DDEVICE,
DEVICE_SYNCHRONIZATION,
COPY_MATRIX_B_FROM_DEVICE_TO_HOST,
FREE_DEVICE_MATRIX_A,
FREE_DEVICE_MATRIX_B
}Cuda_Event_T;

typedef enum Corner_Cell_Name_Tag
{
LEFT_TOP_PADDING_CORNER,
RIGHT_TOP_PADDING_CORNER,
LEFT_BOTTOM_PADDING_CORNER,
RIGHT_BOTTOM_PADDING_CORNER,
NUM_OF_CORNERS
}Corner_Cell_Name_T;

typedef enum Ver_Side_Cell_Name_Tag
{
LEFT_PADDING_CELL,
RIGHT_PADDING_CELL,
NUM_OF_VER_SIDES
}Ver_Side_Cell_Name_T;

typedef enum Hor_Side_Cell_Name_Tag
{
TOP_PADDING_CELL,
BOTTOM_PADDING_CELL,
NUM_OF_HOR_SIDES
}Hor_Side_Cell_Name_T;

/***********************************************************************************\
* Structures
\***********************************************************************************/
typedef struct Result_Tag
{
Status_T status;
int index;
}Result_T;

typedef struct Cell_Tag
{
int r_idx; /* Row index */
int c_idx; /* Column index */
}Cell_T;

/***********************************************************************************\
* Function Macros
\***********************************************************************************/
/* Convert matrix index to array index */
#define MATRIX_TO_ARRAY_INDEX(r_idx, c_idx, num_cols) ((r_idx*num_cols) + c_idx)

/***********************************************************************************\
* CUDA Kernel Device code for 2D Convolution
\***********************************************************************************/

/***********************************************************************************\
* Host code for 2D Convolution and comparing the result with device 2D Convolution
\***********************************************************************************/
__global__ void conv2DDevice(const int *in,  int *out)
{
int g_col_idx = blockDim.x * blockIdx.x + threadIdx.x;
int g_row_idx = blockDim.y * blockIdx.y + threadIdx.y;
int l_col_idx = threadIdx.x + KERNEL_SIZE;
int l_row_idx = threadIdx.y + KERNEL_SIZE;

__shared__ int local[NUM_OF_LOCAL_ROWS*NUM_OF_LOCAL_COLS];

/* Convert from matrix indexing to array indexing */
int g_idx = MATRIX_TO_ARRAY_INDEX(g_row_idx, g_col_idx, NUM_OF_GLOBAL_COLS);
int l_idx = MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx, NUM_OF_LOCAL_COLS);

if ((g_row_idx < NUM_OF_GLOBAL_ROWS) && (g_col_idx < NUM_OF_GLOBAL_COLS))
{
/* Read input elements into shared memory */

/* Fill the internal (BLOCK_SIZE*BLOCK_SIZE) matrix */
local[l_idx] = in[g_idx];

/* Fill the left and right padding columns of local memory */
if (threadIdx.x < KERNEL_SIZE)
{
Cell_T l_ver_side[NUM_OF_VER_SIDES];
Cell_T g_ver_side[NUM_OF_VER_SIDES];

/* Find left and right padding column indices of local memory */
l_ver_side[LEFT_PADDING_CELL].r_idx = l_row_idx;
l_ver_side[LEFT_PADDING_CELL].c_idx = l_col_idx - KERNEL_SIZE;

l_ver_side[RIGHT_PADDING_CELL].r_idx = l_row_idx;
l_ver_side[RIGHT_PADDING_CELL].c_idx = l_col_idx + BLOCK_SIZE;

/* Find indices of global memory whose data needs to be filled
into the left and right padding columns of local memory */
g_ver_side[LEFT_PADDING_CELL].r_idx = g_row_idx;
g_ver_side[LEFT_PADDING_CELL].c_idx = g_col_idx - KERNEL_SIZE;

g_ver_side[RIGHT_PADDING_CELL].r_idx = g_row_idx;
g_ver_side[RIGHT_PADDING_CELL].c_idx = g_col_idx + BLOCK_SIZE;

for (int cell = LEFT_PADDING_CELL; cell < NUM_OF_VER_SIDES; ++cell)
{
bool within_bounds = FALSE;

/* Check if the cell is within bounds of global matrix */
if (LEFT_PADDING_CELL == cell) {
within_bounds = (g_ver_side[cell].c_idx >= 0);
}

if (RIGHT_PADDING_CELL == cell) {
within_bounds = (g_ver_side[cell].c_idx < NUM_OF_GLOBAL_COLS);
}

/* Copy corner into local memory if it is within the bounds of global matrix */
/* Convert from matrix indexing to array indexing */
int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_ver_side[cell].r_idx, l_ver_side[cell].c_idx, NUM_OF_LOCAL_COLS);
int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_ver_side[cell].r_idx, g_ver_side[cell].c_idx, NUM_OF_GLOBAL_COLS);
if (TRUE == within_bounds) {
local[pad_l_idx] = in[pad_g_idx];
}
else {
local[pad_l_idx] = 0;
}
}
}

/* Fill the top and bottom padding rows */
if (threadIdx.y < KERNEL_SIZE)
{
Cell_T l_hor_side[NUM_OF_HOR_SIDES];
Cell_T g_hor_side[NUM_OF_HOR_SIDES];

/* Find top and bottom padding row indices of local memory */
l_hor_side[TOP_PADDING_CELL].r_idx = l_row_idx - KERNEL_SIZE;
l_hor_side[TOP_PADDING_CELL].c_idx = l_col_idx;

l_hor_side[BOTTOM_PADDING_CELL].r_idx = l_row_idx + BLOCK_SIZE;
l_hor_side[BOTTOM_PADDING_CELL].c_idx = l_col_idx;

/* Find indices of global memory whose data needs to be filled
into the top and bottom padding rows of local memory */
g_hor_side[TOP_PADDING_CELL].r_idx = g_row_idx - KERNEL_SIZE;
g_hor_side[TOP_PADDING_CELL].c_idx = g_col_idx;

g_hor_side[BOTTOM_PADDING_CELL].r_idx = g_row_idx + BLOCK_SIZE;
g_hor_side[BOTTOM_PADDING_CELL].c_idx = g_col_idx;

for (int cell = TOP_PADDING_CELL; cell < NUM_OF_HOR_SIDES; ++cell)
{
bool within_bounds = FALSE;

/* Check if the cell is within bounds of global matrix */
if (TOP_PADDING_CELL == cell) {
within_bounds = (g_hor_side[cell].r_idx >= 0);
}

if (BOTTOM_PADDING_CELL == cell) {
within_bounds = (g_hor_side[cell].r_idx < NUM_OF_GLOBAL_ROWS);
}

/* Copy corner into local memory if it is within the bounds of global matrix */
/* Convert from matrix indexing to array indexing */
int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_hor_side[cell].r_idx, l_hor_side[cell].c_idx, NUM_OF_LOCAL_COLS);
int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_hor_side[cell].r_idx, g_hor_side[cell].c_idx, NUM_OF_GLOBAL_COLS);
if (TRUE == within_bounds) {
local[pad_l_idx] = in[pad_g_idx];
}
else {
local[pad_l_idx] = 0;
}
}
}

/* Fill the corners */
if ((threadIdx.x) < KERNEL_SIZE && (threadIdx.y < KERNEL_SIZE))
{
Cell_T l_corner[NUM_OF_CORNERS];
Cell_T g_corner[NUM_OF_CORNERS];

/* Find left top, right top, left bottom and right bottom padding
corner indices of local memory */
l_corner[LEFT_TOP_PADDING_CORNER].r_idx = l_row_idx - KERNEL_SIZE;
l_corner[LEFT_TOP_PADDING_CORNER].c_idx = l_col_idx - KERNEL_SIZE;

l_corner[RIGHT_TOP_PADDING_CORNER].r_idx = l_row_idx - KERNEL_SIZE;
l_corner[RIGHT_TOP_PADDING_CORNER].c_idx = l_col_idx + BLOCK_SIZE;

l_corner[LEFT_BOTTOM_PADDING_CORNER].r_idx = l_row_idx + BLOCK_SIZE;
l_corner[LEFT_BOTTOM_PADDING_CORNER].c_idx = l_col_idx - KERNEL_SIZE;

l_corner[RIGHT_BOTTOM_PADDING_CORNER].r_idx = l_row_idx + BLOCK_SIZE;
l_corner[RIGHT_BOTTOM_PADDING_CORNER].c_idx = l_col_idx + BLOCK_SIZE;

/* Find indices of global memory whose data needs to be filled
into the left top, right top, left bottom and right bottom padding
corners of local memory */
g_corner[LEFT_TOP_PADDING_CORNER].r_idx = g_row_idx - KERNEL_SIZE;
g_corner[LEFT_TOP_PADDING_CORNER].c_idx = g_col_idx - KERNEL_SIZE;

g_corner[RIGHT_TOP_PADDING_CORNER].r_idx = g_row_idx - KERNEL_SIZE;
g_corner[RIGHT_TOP_PADDING_CORNER].c_idx = g_col_idx + BLOCK_SIZE;

g_corner[LEFT_BOTTOM_PADDING_CORNER].r_idx = g_row_idx + BLOCK_SIZE;
g_corner[LEFT_BOTTOM_PADDING_CORNER].c_idx = g_col_idx - KERNEL_SIZE;

g_corner[RIGHT_BOTTOM_PADDING_CORNER].r_idx = g_row_idx + BLOCK_SIZE;
g_corner[RIGHT_BOTTOM_PADDING_CORNER].c_idx = g_col_idx + BLOCK_SIZE;

for (int corner = LEFT_TOP_PADDING_CORNER; corner < NUM_OF_CORNERS; ++corner)
{
bool within_bounds = FALSE;

/* Check if the corner is within bounds of global matrix */
if (LEFT_TOP_PADDING_CORNER == corner){
within_bounds = ((g_corner[corner].r_idx >= 0) && (g_corner[corner].c_idx >= 0));
}

if (RIGHT_TOP_PADDING_CORNER == corner){
within_bounds = ((g_corner[corner].r_idx >= 0) && (g_corner[corner].c_idx < NUM_OF_GLOBAL_COLS));
}

if (LEFT_BOTTOM_PADDING_CORNER == corner){
within_bounds = ((g_corner[corner].r_idx < NUM_OF_GLOBAL_ROWS) && (g_corner[corner].c_idx >= 0));
}

if (RIGHT_BOTTOM_PADDING_CORNER == corner){
within_bounds = ((g_corner[corner].r_idx < NUM_OF_GLOBAL_ROWS) && (g_corner[corner].c_idx < NUM_OF_GLOBAL_COLS));
}

/* Copy corner into local memory if it is within the bounds of global matrix */
/* Convert from matrix indexing to array indexing */
int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_corner[corner].r_idx, l_corner[corner].c_idx, NUM_OF_LOCAL_COLS);
int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_corner[corner].r_idx, g_corner[corner].c_idx, NUM_OF_GLOBAL_COLS);
if (TRUE == within_bounds){
local[pad_l_idx] = in[pad_g_idx];
}
else {
local[pad_l_idx] = 0;
}
}
}

__syncthreads();

/* Apply convolution */
int result = 0;
for (int row_offset = -KERNEL_SIZE; row_offset <= KERNEL_SIZE; ++row_offset)
{
for (int col_offset = -KERNEL_SIZE; col_offset <= KERNEL_SIZE; ++col_offset)
{
/* Convert local matrix row and column to local element index */
int l_ele_idx = MATRIX_TO_ARRAY_INDEX((l_row_idx + row_offset), (l_col_idx + col_offset), NUM_OF_LOCAL_COLS);
result += local[l_ele_idx];
}
}

/* Store the result */
out[g_idx] = result;
}
}