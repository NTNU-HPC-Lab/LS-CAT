#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize



//typedef float float;
float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数
//BOOL elitism = TRUE; //是否精英选择

__global__ void mutationFcn(float *populationArray, hiprandState_t *states) {
//printf("mutationFcn\n");
int idx = threadIdx.x;
hiprandState_t s = states[idx];
hiprandState_t t = states[idx];
float ss = hiprand_uniform(&s);
int tt = hiprand(&t);

float scale = 0.5, shrink = 0.75;
scale -= scale * shrink * idx / maxGeneration;

//判断当前个体是否变异
if (ss < mutationRate){
for (int j = 0; j < chromosomeSize; j++) {
//判断当前染色体是否变异
if (tt % 2 != 0) {
float tmpChromosome;
do {
tmpChromosome = populationArray[idx * chromosomeSize + j] + scale * (c_UB[j] - c_LB[j]) * ss;
//判断是否越界
} while (tmpChromosome > c_UB[j] || tmpChromosome < c_LB[j]);
populationArray[idx * chromosomeSize + j] = tmpChromosome;
}
}
}
}