#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize



//typedef float float;
float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数
//BOOL elitism = TRUE; //是否精英选择

__global__ void selectPre(float *fitness, float *Fitness, float *tmpFitness, float *populationArray, float *tmpPopulationArray){
int idx = threadIdx.x;
Fitness[idx] = 1 / fitness[idx];
__syncthreads();
tmpFitness[idx] = fitness[idx];
__syncthreads();
for(int i = 0; i < chromosomeSize; i++){
tmpPopulationArray[idx * chromosomeSize + i] = populationArray[idx * chromosomeSize + i];
}
__syncthreads();
}