#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize



//typedef float float;
float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数
//BOOL elitism = TRUE; //是否精英选择

__global__ void selectFcn(float *populationArray, float *tmpPopulationArray, float *fitness, float *Fitness, float *tmpFitness, float sumFitness, float *populationPro, hiprandState_t *states) {
//printf("selectFcn\n");
int idx = threadIdx.x;

//每个个体被选择的概率
populationPro[idx] = Fitness[idx] / sumFitness;
__syncthreads();

//轮盘选择
int index;
hiprandState_t s;
s = states[idx];
float ss = hiprand_uniform(&s);
while (ss < 0.0001)
ss = hiprand_uniform(&s);
//printf("%e\n", ss);

for (int j = 0; j < populationSize; j++) {
ss -= populationPro[j];
if (ss <= 0) {
index = j;
//printf("%d\n", index);
break;
}
}

//产生新种群
for (int j = 0; j < chromosomeSize; j++) {
populationArray[idx * chromosomeSize + j] = tmpPopulationArray[index * chromosomeSize + j];
}
__syncthreads();
fitness[idx] = tmpFitness[index];
__syncthreads();
}