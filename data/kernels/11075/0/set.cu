#include "hip/hip_runtime.h"
#include "includes.h"

#define imin(a,b) (a<b?a:b)
__global__ void set(double *dx,int N)
{
int tid=threadIdx.x+blockIdx.x*blockDim.x ;
if (tid<N)
dx[tid]=0.0;
}