#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyNumbersGPU(float *pDataA, float *pDataB, float *pResult)
{
int tid = (blockIdx.y * 128 * 256) + blockIdx.x * 256 + threadIdx.x;
pResult[tid] = sqrt(pDataA[tid] * pDataB[tid] / 12.34567) * sin(pDataA[tid]);

}