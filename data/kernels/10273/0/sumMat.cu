#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMat(double *A, double *B, double *C, int N)
{
int col = blockDim.x*blockIdx.x + threadIdx.x;
int row = blockDim.y*blockIdx.y + threadIdx.y;

if( (col < N) && (row < N)){
C[col*N + row] = A[col*N + row] + B[col*N + row];
//C[col][row] = B[col][row] + A[col][row];
}

}