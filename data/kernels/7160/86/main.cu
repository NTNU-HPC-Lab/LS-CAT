#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computeCovDxdPi.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int *starting_voxel_id = NULL;
hipMalloc(&starting_voxel_id, XSIZE*YSIZE);
int *voxel_id = NULL;
hipMalloc(&voxel_id, XSIZE*YSIZE);
int valid_points_num = 1;
double *inverse_covariance = NULL;
hipMalloc(&inverse_covariance, XSIZE*YSIZE);
int voxel_num = 1;
double gauss_d1 = 1;
double gauss_d2 = 1;
double *point_gradients = NULL;
hipMalloc(&point_gradients, XSIZE*YSIZE);
double *cov_dxd_pi = NULL;
hipMalloc(&cov_dxd_pi, XSIZE*YSIZE);
int valid_voxel_num = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeCovDxdPi<<<gridBlock,threadBlock>>>(valid_points,starting_voxel_id,voxel_id,valid_points_num,inverse_covariance,voxel_num,gauss_d1,gauss_d2,point_gradients,cov_dxd_pi,valid_voxel_num);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computeCovDxdPi<<<gridBlock,threadBlock>>>(valid_points,starting_voxel_id,voxel_id,valid_points_num,inverse_covariance,voxel_num,gauss_d1,gauss_d2,point_gradients,cov_dxd_pi,valid_voxel_num);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computeCovDxdPi<<<gridBlock,threadBlock>>>(valid_points,starting_voxel_id,voxel_id,valid_points_num,inverse_covariance,voxel_num,gauss_d1,gauss_d2,point_gradients,cov_dxd_pi,valid_voxel_num);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}