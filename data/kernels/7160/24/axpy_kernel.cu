#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}