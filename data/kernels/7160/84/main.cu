#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computeExCovX.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *trans_x = NULL;
hipMalloc(&trans_x, XSIZE*YSIZE);
float *trans_y = NULL;
hipMalloc(&trans_y, XSIZE*YSIZE);
float *trans_z = NULL;
hipMalloc(&trans_z, XSIZE*YSIZE);
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int *starting_voxel_id = NULL;
hipMalloc(&starting_voxel_id, XSIZE*YSIZE);
int *voxel_id = NULL;
hipMalloc(&voxel_id, XSIZE*YSIZE);
int valid_points_num = 1;
double *centr_x = NULL;
hipMalloc(&centr_x, XSIZE*YSIZE);
double *centr_y = NULL;
hipMalloc(&centr_y, XSIZE*YSIZE);
double *centr_z = NULL;
hipMalloc(&centr_z, XSIZE*YSIZE);
double gauss_d1 = 1;
double gauss_d2 = 1;
double *e_x_cov_x = NULL;
hipMalloc(&e_x_cov_x, XSIZE*YSIZE);
double *icov00 = NULL;
hipMalloc(&icov00, XSIZE*YSIZE);
double *icov01 = NULL;
hipMalloc(&icov01, XSIZE*YSIZE);
double *icov02 = NULL;
hipMalloc(&icov02, XSIZE*YSIZE);
double *icov10 = NULL;
hipMalloc(&icov10, XSIZE*YSIZE);
double *icov11 = NULL;
hipMalloc(&icov11, XSIZE*YSIZE);
double *icov12 = NULL;
hipMalloc(&icov12, XSIZE*YSIZE);
double *icov20 = NULL;
hipMalloc(&icov20, XSIZE*YSIZE);
double *icov21 = NULL;
hipMalloc(&icov21, XSIZE*YSIZE);
double *icov22 = NULL;
hipMalloc(&icov22, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeExCovX<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centr_x,centr_y,centr_z,gauss_d1,gauss_d2,e_x_cov_x,icov00,icov01,icov02,icov10,icov11,icov12,icov20,icov21,icov22);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computeExCovX<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centr_x,centr_y,centr_z,gauss_d1,gauss_d2,e_x_cov_x,icov00,icov01,icov02,icov10,icov11,icov12,icov20,icov21,icov22);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computeExCovX<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centr_x,centr_y,centr_z,gauss_d1,gauss_d2,e_x_cov_x,icov00,icov01,icov02,icov10,icov11,icov12,icov20,icov21,icov22);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}