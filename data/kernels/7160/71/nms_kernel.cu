#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float devIoU(float const *const a, float const *const b)
{
float left = max(a[0], b[0]), right = min(a[2], b[2]);
float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
float interS = width * height;
float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
return interS / (Sa + Sb - interS);
}
__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh, const float *dev_boxes, unsigned long long *dev_mask, const int NUM_BOX_CORNERS)
{
const int row_start = blockIdx.y;
const int col_start = blockIdx.x;

const int block_threads = blockDim.x;

const int row_size =
min(n_boxes - row_start * block_threads, block_threads);
const int col_size =
min(n_boxes - col_start * block_threads, block_threads);

__shared__ float block_boxes[NUM_THREADS_MACRO * NUM_2D_BOX_CORNERS_MACRO];
if (threadIdx.x < col_size)
{
block_boxes[threadIdx.x * NUM_BOX_CORNERS + 0] = dev_boxes[(block_threads * col_start + threadIdx.x) * NUM_BOX_CORNERS + 0];
block_boxes[threadIdx.x * NUM_BOX_CORNERS + 1] = dev_boxes[(block_threads * col_start + threadIdx.x) * NUM_BOX_CORNERS + 1];
block_boxes[threadIdx.x * NUM_BOX_CORNERS + 2] = dev_boxes[(block_threads * col_start + threadIdx.x) * NUM_BOX_CORNERS + 2];
block_boxes[threadIdx.x * NUM_BOX_CORNERS + 3] = dev_boxes[(block_threads * col_start + threadIdx.x) * NUM_BOX_CORNERS + 3];
}
__syncthreads();

if (threadIdx.x < row_size)
{
const int cur_box_idx = block_threads * row_start + threadIdx.x;
const float cur_box[NUM_2D_BOX_CORNERS_MACRO] = {dev_boxes[cur_box_idx*NUM_BOX_CORNERS + 0],
dev_boxes[cur_box_idx*NUM_BOX_CORNERS + 1],
dev_boxes[cur_box_idx*NUM_BOX_CORNERS + 2],
dev_boxes[cur_box_idx*NUM_BOX_CORNERS + 3]};
unsigned long long t = 0;
int start = 0;
if (row_start == col_start)
{
start = threadIdx.x + 1;
}
for (int i = start; i < col_size; i++)
{
if (devIoU(cur_box, block_boxes + i * NUM_BOX_CORNERS) > nms_overlap_thresh)
{
t |= 1ULL << i;
}
}
const int col_blocks = DIVUP(n_boxes, block_threads);
dev_mask[cur_box_idx * col_blocks + col_start] = t;
}
}