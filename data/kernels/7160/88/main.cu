#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computeHessianListS1.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *trans_x = NULL;
hipMalloc(&trans_x, XSIZE*YSIZE);
float *trans_y = NULL;
hipMalloc(&trans_y, XSIZE*YSIZE);
float *trans_z = NULL;
hipMalloc(&trans_z, XSIZE*YSIZE);
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int *starting_voxel_id = NULL;
hipMalloc(&starting_voxel_id, XSIZE*YSIZE);
int *voxel_id = NULL;
hipMalloc(&voxel_id, XSIZE*YSIZE);
int valid_points_num = 1;
double *centroid_x = NULL;
hipMalloc(&centroid_x, XSIZE*YSIZE);
double *centroid_y = NULL;
hipMalloc(&centroid_y, XSIZE*YSIZE);
double *centroid_z = NULL;
hipMalloc(&centroid_z, XSIZE*YSIZE);
double gauss_d1 = 1;
double gauss_d2 = 1;
double *hessians = NULL;
hipMalloc(&hessians, XSIZE*YSIZE);
double *e_x_cov_x = NULL;
hipMalloc(&e_x_cov_x, XSIZE*YSIZE);
double *tmp_hessian = NULL;
hipMalloc(&tmp_hessian, XSIZE*YSIZE);
double *cov_dxd_pi = NULL;
hipMalloc(&cov_dxd_pi, XSIZE*YSIZE);
double *point_gradients = NULL;
hipMalloc(&point_gradients, XSIZE*YSIZE);
int valid_voxel_num = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeHessianListS1<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,gauss_d1,gauss_d2,hessians,e_x_cov_x,tmp_hessian,cov_dxd_pi,point_gradients,valid_voxel_num);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computeHessianListS1<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,gauss_d1,gauss_d2,hessians,e_x_cov_x,tmp_hessian,cov_dxd_pi,point_gradients,valid_voxel_num);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computeHessianListS1<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,gauss_d1,gauss_d2,hessians,e_x_cov_x,tmp_hessian,cov_dxd_pi,point_gradients,valid_voxel_num);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}