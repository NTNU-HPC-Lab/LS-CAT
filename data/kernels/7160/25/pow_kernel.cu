#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}