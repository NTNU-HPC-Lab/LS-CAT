#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sort_boxes_by_indexes_kernel(float* filtered_box, int* filtered_dir, float* box_for_nms, int* indexes, int filter_count, float* sorted_filtered_boxes, int* sorted_filtered_dir, float* sorted_box_for_nms, const int NUM_BOX_CORNERS, const int NUM_OUTPUT_BOX_FEATURE)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if(tid < filter_count)
{
int sort_index = indexes[tid];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 0] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 0];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 1] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 1];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 2] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 2];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 3] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 3];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 4] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 4];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 5] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 5];
sorted_filtered_boxes[tid*NUM_OUTPUT_BOX_FEATURE + 6] = filtered_box[sort_index*NUM_OUTPUT_BOX_FEATURE + 6];

sorted_filtered_dir[tid] = filtered_dir[sort_index];


sorted_box_for_nms[tid*NUM_BOX_CORNERS + 0] = box_for_nms[sort_index*NUM_BOX_CORNERS + 0];
sorted_box_for_nms[tid*NUM_BOX_CORNERS + 1] = box_for_nms[sort_index*NUM_BOX_CORNERS + 1];
sorted_box_for_nms[tid*NUM_BOX_CORNERS + 2] = box_for_nms[sort_index*NUM_BOX_CORNERS + 2];
sorted_box_for_nms[tid*NUM_BOX_CORNERS + 3] = box_for_nms[sort_index*NUM_BOX_CORNERS + 3];
}
}