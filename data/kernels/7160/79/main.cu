#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computePointHessian0.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
float *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
float *z = NULL;
hipMalloc(&z, XSIZE*YSIZE);
int points_num = 1;
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int valid_points_num = 1;
double *dh_ang = NULL;
hipMalloc(&dh_ang, XSIZE*YSIZE);
double *ph93 = NULL;
hipMalloc(&ph93, XSIZE*YSIZE);
double *ph103 = NULL;
hipMalloc(&ph103, XSIZE*YSIZE);
double *ph113 = NULL;
hipMalloc(&ph113, XSIZE*YSIZE);
double *ph123 = NULL;
hipMalloc(&ph123, XSIZE*YSIZE);
double *ph94 = NULL;
hipMalloc(&ph94, XSIZE*YSIZE);
double *ph133 = NULL;
hipMalloc(&ph133, XSIZE*YSIZE);
double *ph104 = NULL;
hipMalloc(&ph104, XSIZE*YSIZE);
double *ph143 = NULL;
hipMalloc(&ph143, XSIZE*YSIZE);
double *ph114 = NULL;
hipMalloc(&ph114, XSIZE*YSIZE);
double *ph153 = NULL;
hipMalloc(&ph153, XSIZE*YSIZE);
double *ph95 = NULL;
hipMalloc(&ph95, XSIZE*YSIZE);
double *ph163 = NULL;
hipMalloc(&ph163, XSIZE*YSIZE);
double *ph105 = NULL;
hipMalloc(&ph105, XSIZE*YSIZE);
double *ph173 = NULL;
hipMalloc(&ph173, XSIZE*YSIZE);
double *ph115 = NULL;
hipMalloc(&ph115, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computePointHessian0<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph93,ph103,ph113,ph123,ph94,ph133,ph104,ph143,ph114,ph153,ph95,ph163,ph105,ph173,ph115);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computePointHessian0<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph93,ph103,ph113,ph123,ph94,ph133,ph104,ph143,ph114,ph153,ph95,ph163,ph105,ph173,ph115);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computePointHessian0<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph93,ph103,ph113,ph123,ph94,ph133,ph104,ph143,ph114,ph153,ph95,ph163,ph105,ph173,ph115);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}