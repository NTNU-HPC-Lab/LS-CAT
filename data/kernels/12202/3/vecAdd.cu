#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(unsigned int *A_d, unsigned int *B_d, unsigned int *C_d, int WORK_SIZE) {

//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// **** Populate vecADD kernel function ****
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < WORK_SIZE)
C_d[id] = A_d[id] + B_d[id];


}