#include "hip/hip_runtime.h"
#include "includes.h"
//---------------------------------------------------------------------------------

//---------------------------------------------------------------------------------
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// **** 	A = M x N		****			AxB=C
//****		B = N x K		****
//**** 	C = M x K		****
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@


static const int M = 3;
static const int N = 5;
static const int K = 4;
static const int TILE_WIDTH = 2;

using namespace std;
//---------------------------------------------------------------------------------
/**
* This macro checks return value of the CUDA runtime call and exits
* the application if the call failed.
*/
__global__ void MatrixMulKernel(int ARows,int ACols, int BRows, int BCols, int CRows, int CCols,unsigned int* A_d, unsigned int *B_d, unsigned int *C_d) {

//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// **** Populate matrixMultiplication kernel function ****
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@


int CValue = 0;

int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

__shared__ int As[TILE_WIDTH][TILE_WIDTH];
__shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

for (int k = 0; k < (TILE_WIDTH + ACols - 1)/TILE_WIDTH; k++) {

if (k*TILE_WIDTH + threadIdx.x < ACols && Row < ARows)
As[threadIdx.y][threadIdx.x] = A_d[Row*ACols + k*TILE_WIDTH + threadIdx.x];
else
As[threadIdx.y][threadIdx.x] = 0;

if (k*TILE_WIDTH + threadIdx.y < BRows && Col < BCols)
Bs[threadIdx.y][threadIdx.x] = B_d[(k*TILE_WIDTH + threadIdx.y)*BCols + Col];
else
Bs[threadIdx.y][threadIdx.x] = 0;

__syncthreads();

for (int n = 0; n < TILE_WIDTH; ++n)
CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

__syncthreads();
}

if (Row < CRows && Col < CCols)
C_d[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
(blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;



}