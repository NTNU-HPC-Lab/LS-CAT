#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTranspose(unsigned int* A_d, unsigned int *T_d, int n) {

//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// **** Populate matrixTranspose kernel function ****
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
int i = threadIdx.x;
int j = threadIdx.y;

if(i<n&&j<n)
T_d[i+j*n] = A_d[j+i*n];

}