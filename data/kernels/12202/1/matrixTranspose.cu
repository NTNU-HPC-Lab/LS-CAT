#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTranspose(unsigned int* A_d, unsigned int *T_d, int rowCount, int colCount) {

//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
// **** Populate vecADD kernel function ****
//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
if (row < rowCount && col < colCount){
T_d[col*rowCount+row] = A_d[row*colCount+col];
}

}