#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVectors( float *d_a, float *d_b, float *d_c, int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
d_c[i] = d_a[i] + d_b[i];
}
}