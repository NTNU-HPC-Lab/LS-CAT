#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1){
int i = threadIdx.x + blockDim.x * blockIdx.x;
int stride = blockDim.x * gridDim.x;

while (i < size1){
atomicAdd(&(histo1[buffer1[i]]),1);
i += stride;
}
}