#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float explicitLocalStepHeat( float unjpo, float unjmo, float unj, float r)
{
return (1 - 2 * r)*unj + r*unjmo + r * unjpo;
}
__global__ void explicitTimestepHeat( int size, float *d_currentVal, float *d_nextVal, float r )
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
if (i < 2)
{
d_nextVal[i] == 0;
}
else if (i > size - 2)
{
d_nextVal[i] == 0;
}
else
{
d_nextVal[i] = explicitLocalStepHeat(
d_currentVal[i + 1],
d_currentVal[i - 1],
d_currentVal[i],
r);
}
}
}