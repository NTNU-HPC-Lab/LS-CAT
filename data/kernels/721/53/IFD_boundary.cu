#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IFD_boundary( int size, double *d_Price, double lambda_U, double lambda_L )
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
if (i == 0)//top condition
{
d_Price[i] = lambda_U;
}

else if (i == size - 1) //bottom condition
{
d_Price[i] = 0.0;
}
}
}