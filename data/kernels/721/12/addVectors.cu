#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVectors( float *d_A, float *d_B, float *d_C, int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
d_C[i] = d_A[i] + d_B[i];
}
}