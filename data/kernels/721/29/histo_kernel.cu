#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1){

// Phase 1 ------------------------------------------------------------
__shared__ unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x + blockDim.x * blockIdx.x;
int stride = blockDim.x * gridDim.x;

while (i < size1){
atomicAdd(&(temp[buffer1[i]]),1);
i += stride;
}
__syncthreads();
//---------------------------------------------------------------------

// Phase 2 ------------------------------------------------------------
atomicAdd(&(histo1[threadIdx.x]), temp[threadIdx.x]);
//---------------------------------------------------------------------
}