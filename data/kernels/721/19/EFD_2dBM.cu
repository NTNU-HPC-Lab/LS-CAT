#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void EFD_2dBM( int width, int height, int pitch_n, int pitch_npo, float *d_val_n, float *d_val_npo, float alpha, float beta ){
int idx = blockIdx.x;	//row
int idy = threadIdx.x;	//column

if ((idx < height) && (idy <width )){
//d_val_npo[i] = Pu * d_val_n[i + 1] + Pm * d_val_n[i] + Pd * d_val_n[i - 1];
d_val_npo[idx*(pitch_npo / sizeof(float)) + idy] = alpha*(d_val_n[(idx+1)*(pitch_n / sizeof(float)) + idy]
+ d_val_n[(idx - 1)*(pitch_n / sizeof(float)) + idy])
+ beta*(d_val_n[idx*(pitch_n / sizeof(float)) + idy+1]
+ d_val_n[idx*(pitch_n / sizeof(float)) + idy-1])
+ (1.0-2.0*alpha-2.0*beta)*d_val_n[idx*(pitch_n / sizeof(float)) + idy];

//modify the ones on the top
if (idx == 0){
d_val_npo[idx*(pitch_npo / sizeof(float)) + idy] = d_val_npo[(idx+1)*(pitch_npo / sizeof(float)) + idy];
}
//modify the ones on the bottom
if (idx == (height-1)){
d_val_npo[idx*(pitch_npo / sizeof(float)) + idy] = d_val_npo[(idx - 1)*(pitch_npo / sizeof(float)) + idy];
}
//modify the ones on the left
if (idy == 0){
d_val_npo[idx*(pitch_npo / sizeof(float)) + idy] = d_val_npo[(idx - 1)*(pitch_npo / sizeof(float)) + idy+1];
}
//modify the ones on the right
if (idx == (width - 1)){
d_val_npo[idx*(pitch_npo / sizeof(float)) + idy] = d_val_npo[(idx - 1)*(pitch_npo / sizeof(float)) + idy-1];
}
}
}