#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addArray( float *d_a, float *d_b, float *d_c, int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i >= size)
{
return;
}
d_c[i] = d_a[i] + d_b[i];
}