#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *x, int n)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
x[i] = sqrt(pow(3.14159, i));
}
}