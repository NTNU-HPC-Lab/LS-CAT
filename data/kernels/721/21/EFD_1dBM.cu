#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void EFD_1dBM( int size, float *d_val_n, float *d_val_npo, float Pu, float Pm, float Pd ){
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
d_val_npo[i] = Pu * d_val_n[i + 1] + Pm * d_val_n[i] + Pd * d_val_n[i - 1];

if (i == 0)
{
d_val_npo[i] = d_val_npo[1];
}
else if (i == size - 1)
{
d_val_npo[i] = d_val_npo[i - 1];
}
}
}