#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modify_i_j( int width, int height, int pitch, float *d_array, int i, int j, float change_to ){
//we want to change the [i,j]-th of the 2-dim array
int idx = blockIdx.x; //row
int idy = threadIdx.x; //column

//we can do index by pointer:
//if ((idx == i) && (idy == j)){
//float* row = (float *)((char*)d_array + idx*pitch);
//	row[idy] = change_to;
//}

//or, a more convenient way is to do index just use idx and idy
if ((idx==i)&&(idy==j))
{
d_array[idx*(pitch/sizeof(float)) + idy] = change_to;
}

}