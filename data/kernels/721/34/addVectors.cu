#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVectors( int size, float *d_a, float *d_b, float *d_c)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid < size)
{
d_c[tid] = d_a[tid] + d_b[tid];
}
}