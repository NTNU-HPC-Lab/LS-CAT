#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int * in, int * out){
int globalid = blockIdx.x*blockDim.x + threadIdx.x;
__shared__ int s_array[BLOCK_DIM];

s_array[threadIdx.x] = in[globalid];
__syncthreads();

for (int i = blockDim.x / 2; i > 0; i /= 2){
if (threadIdx.x < i){
s_array[threadIdx.x] += s_array[threadIdx.x + i];
}
__syncthreads();
}

if (threadIdx.x == 0)
out[blockIdx.x] = s_array[0];
}