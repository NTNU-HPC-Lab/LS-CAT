#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void marks(float * media, int * final){
int thread = blockIdx.x*blockDim.x + threadIdx.x;
final[thread] =	(media[thread] == (int)media[thread]) * (int)media[thread] +
(media[thread] != (int)media[thread] && media[thread] > 4 && media[thread] < 5)* 4 +
(media[thread] != (int)media[thread] && media[thread] > 9)* 9 +
(media[thread] != (int)media[thread] && (media[thread] < 4 || (media[thread] > 5 && media[thread] < 9))) * ((int)media[thread] + 1);
}