#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int * A, int * B, int * C){
int thread = blockIdx.x*blockDim.x + threadIdx.x;
C[thread] = A[thread] + B[thread];
}