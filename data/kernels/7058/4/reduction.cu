#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int * in, int * out){
int globalid = blockIdx.x*blockDim.x + threadIdx.x;
__shared__ int s_array[BLOCK_DIM];

s_array[threadIdx.x] = in[globalid];
__syncthreads();

for (int i = 1; i < blockDim.x; i *= 2){
if (threadIdx.x % (2*i) == 0){
s_array[threadIdx.x] += s_array[threadIdx.x+i];
}
__syncthreads();
}

if (threadIdx.x == 0)
out[blockIdx.x] = s_array[0];
}