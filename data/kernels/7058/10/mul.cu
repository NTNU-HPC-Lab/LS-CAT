#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul(int * A, int * B, int * C){
int i = blockIdx.x;
int j = threadIdx.x;
C[i * N + j] = 0;
for (int k = 0; k < N; k++){
C[i * N + j] += A[i * N + k] * B[k * N + j];
}
}