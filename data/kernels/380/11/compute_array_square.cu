#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_array_square(float* array, float* outArray, int size)
{
int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
int num_threads = blockDim.x * gridDim.x;

for(int i = 0; i < size; i += num_threads)
{
int index = i + thread_index;
if(index < size)
{
outArray[index] = array[index] * array[index];
}
}
}