#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FillOnes(float* vec, int value)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx > value) return ;

vec[idx] = 1.0f;
}