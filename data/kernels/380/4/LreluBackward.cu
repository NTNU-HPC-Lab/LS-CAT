#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LreluBackward(float* srcDiff, float* dstDiff, float* srcData, int data_size)
{
int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
int num_threads = blockDim.x * gridDim.x;

for(int i = 0; i < data_size; i += num_threads)
{
int index = i + thread_index;
if(index < data_size)
{
dstDiff[index] = srcDiff[index] * ((srcData[index] > 0) + (srcData[index] <= 0) * 0.01);
}
}

}