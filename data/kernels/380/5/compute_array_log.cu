#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_array_log(float* array, float* groundTruth, int size)
{
int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
int num_threads = blockDim.x * gridDim.x;

for(int i = 0; i < size; i += num_threads)
{
int index = i + thread_index;
if(index < size)
{
groundTruth[index] = log(array[index]) * groundTruth[index];
}
}
}