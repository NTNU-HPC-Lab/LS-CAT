#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dropout_train(float* data, float* outputPtr, int size, float probability)
{
int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
int num_threads = blockDim.x * gridDim.x;
for(int  i = 0; i < size; i += num_threads)
{
int index = i + thread_index;
if(index < size)
{
if(outputPtr[index] < probability)
data[index] = 0;
}
}
}