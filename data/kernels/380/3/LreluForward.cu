#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LreluForward(float* srcData, float* dstData, int data_size)
{
int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
int num_threads = blockDim.x * gridDim.x;
for(int i = 0; i < data_size; i += num_threads)
{
int index = i + thread_index;
if(index < data_size)
{
dstData[index] = srcData[index] > 0 ? srcData[index] : srcData[index] * 0.01;
}
}

}