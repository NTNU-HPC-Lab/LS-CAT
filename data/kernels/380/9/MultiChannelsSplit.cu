#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiChannelsSplit(float* inputs, float* outputs, int outChannels, int offset, int row, int inChannels)
{
int  batchId = blockIdx.x;
float* input = inputs + batchId * inChannels * row * row + offset;

int blockDo  = outChannels * row * row;
for(int i = 0; i < blockDo; i += blockDim.x)
{
int j = i + threadIdx.x;
if(j < blockDo)
{
int pos = batchId * outChannels * row * row;
outputs[pos + j] = input[j];
}
}
}