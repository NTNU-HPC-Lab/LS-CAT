#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
*(output + blockIdx.x * N + i) = -col_sum[i];
}
}