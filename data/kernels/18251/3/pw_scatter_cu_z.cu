#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

const int igpt =
(gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

if (igpt < ngpts) {
c[2 * ghatmap[igpt]] = scale * pwcc[2 * igpt];
c[2 * ghatmap[igpt] + 1] = scale * pwcc[2 * igpt + 1];
if (nmaps == 2) {
c[2 * ghatmap[igpt + ngpts]] = scale * pwcc[2 * igpt];
c[2 * ghatmap[igpt + ngpts] + 1] = -scale * pwcc[2 * igpt + 1];
}
}
}