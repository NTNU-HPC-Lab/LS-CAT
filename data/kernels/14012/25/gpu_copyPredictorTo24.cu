#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo24(int32_t * in, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{
uint8_t * op = out + (index * theOutputPacketBytes);
int32_t	val = (in + index * frameLength)[z];
op += (stride * 3 * z);

op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
}
}