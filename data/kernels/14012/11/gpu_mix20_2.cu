#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix20_2(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
int z = threadIdx.x + blockIdx.x * blockDim.x;
if (z < numSamples)
{
int32_t		l, r;


ip += 3 * z;
ip += (stride - 1) * 3 * z;
l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
u[z] = (l << 8) >> 12;

ip += 3;
r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
v[z] = (r << 8) >> 12;
}
}