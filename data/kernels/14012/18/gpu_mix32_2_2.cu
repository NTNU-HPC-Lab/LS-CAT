#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix32_2_2(int64_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, uint32_t mask, int32_t shift)
{
int z = threadIdx.x + blockIdx.x * blockDim.x;
if (z < numSamples)
{
int32_t		l, r;
int32_t k = z * 2;


int64_t temp = ip[z];


l = (int32_t)temp;
r = temp >> 32;

shiftUV[k + 0] = (uint16_t)(l & mask);
shiftUV[k + 1] = (uint16_t)(r & mask);

l >>= shift;
r >>= shift;

u[z] = l;
v[z] = r;
}
}