#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo32Shift(int32_t * in, uint16_t * shift, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{

int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{
int32_t * op = out + (index * theOutputPacketBytes) / 4;
int32_t	shiftVal = bytesShifted * 8;

//Assert( bytesShifted != 0 );

op += stride * z;
op[0] = ((in + index * frameLength)[z] << shiftVal) | (uint32_t)(shift + index * frameLength * 2)[z];

}
}