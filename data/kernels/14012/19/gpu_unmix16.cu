#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_unmix16(int32_t * u, int32_t * v, int16_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{

int32_t		l, r;
int16_t * op = out + (index * theOutputPacketBytes) / 2;

if (mixres[index] != 0)
{
/* matrixed stereo */
l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
r = l - (v + index * frameLength)[z];
op += stride* z;
op[0] = (int16_t)l;
op[1] = (int16_t)r;
}
else
{
/* Conventional separated stereo. */
op += stride * z;
op[0] = (int16_t)(u + index * frameLength)[z];
op[1] = (int16_t)(v + index * frameLength)[z];
}

}
}