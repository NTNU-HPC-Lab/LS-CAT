#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix24_2_1(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, uint32_t mask, int32_t shift)
{
int z = threadIdx.x + blockIdx.x * blockDim.x;
if (z < numSamples)
{
int32_t		l, r;
int32_t k = z * 2;


ip += 3 * z;
ip += (stride - 1) * 3 * z;
l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
l = (l << 8) >> 8;

ip += 3;
r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
r = (r << 8) >> 8;

shiftUV[k + 0] = (uint16_t)(l & mask);
shiftUV[k + 1] = (uint16_t)(r & mask);

l >>= shift;
r >>= shift;

u[z] = l;
v[z] = r;
}
}