#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_unmix32(int32_t * u, int32_t * v, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, uint16_t * shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;
if (z < numSamples[index])
{
int32_t			shift = bytesShifted * 8;
int32_t		l, r;
int32_t k = z * 2;
int32_t * op = out + (index * theOutputPacketBytes) / 4;

if (mixres[index] != 0)
{
//Assert( bytesShifted != 0 );

/* matrixed stereo with shift */
int32_t		lt, rt;

lt = (u + index * frameLength)[z];
rt = (v + index * frameLength)[z];

l = lt + rt - ((mixres[index] * rt) >> mixbits[index]);
r = l - rt;

op += stride * z;
op[0] = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
op[1] = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];
}
else
{
/* interleaving with shift */
op += stride * z;
op[0] = ((u + index * frameLength)[z] << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
op[1] = ((v + index * frameLength)[z] << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

}
}
}