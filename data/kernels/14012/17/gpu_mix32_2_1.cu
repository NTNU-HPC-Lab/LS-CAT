#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix32_2_1(int64_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
int z = threadIdx.x + blockIdx.x * blockDim.x;
if (z < numSamples)
{
int64_t temp = ip[z];

u[z] = (int32_t)temp;
v[z] = temp >> 32;
}
}