#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo24Shift(int32_t * in, uint16_t * shift, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{

int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{
uint8_t * op = out + (index * theOutputPacketBytes);
int32_t	shiftVal = bytesShifted * 8;

//Assert( bytesShifted != 0 );

int32_t	val = (in + index * frameLength)[z];

val = (val << shiftVal) | (uint32_t)(shift + index * frameLength * 2)[z];


op += (stride * 3 * z);
op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
}
}