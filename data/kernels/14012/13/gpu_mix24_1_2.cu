#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix24_1_2(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t mixres, int32_t m2, int32_t mixbits)
{
int z = threadIdx.x + blockIdx.x * blockDim.x;
if (z < numSamples)
{
int32_t		l, r;

ip += 3 * z;
ip += (stride - 1) * 3 * z;
l = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
l = (l << 8) >> 8;

ip += 3;
r = (int32_t)(((uint32_t)ip[HBYTE] << 16) | ((uint32_t)ip[MBYTE] << 8) | (uint32_t)ip[LBYTE]);
r = (r << 8) >> 8;

u[z] = (mixres * l + m2 * r) >> mixbits;
v[z] = l - r;
}
}