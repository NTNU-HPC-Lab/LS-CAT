#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_unmix24(int32_t * u, int32_t * v, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, uint16_t * shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{

int32_t			shift = bytesShifted * 8;
int32_t		l, r;
int32_t k = z * 2;
uint8_t * op = out + (index * theOutputPacketBytes);

if (mixres[index] != 0)
{
/* matrixed stereo */
l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
r = l - (v + index * frameLength)[z];

l = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
r = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

op += 3 * z;
op += (stride - 1) * 3 * z;
op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
op += 3;

op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
}
else
{
/* Conventional separated stereo. */
l = (u + index * frameLength)[z];
r = (v + index * frameLength)[z];

l = (l << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 0];
r = (r << shift) | (uint32_t)(shiftUV + index * frameLength * 2)[k + 1];

op += 3 * z;
op += (stride - 1) * 3 * z;
op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
op += 3;

op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
}
}
}