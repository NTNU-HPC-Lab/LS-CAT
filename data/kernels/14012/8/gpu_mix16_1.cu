#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix16_1(int32_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t m2, int32_t mixbits, int32_t mixres)
{

int z = threadIdx.x + blockIdx.x * blockDim.x;

if (z < numSamples){

int32_t temp = ip[z];
int32_t		l, r;
l = (int16_t)temp;
r = temp >> 16;
u[z] = (mixres * l + m2 * r) >> mixbits;
v[z] = l - r;
}
}