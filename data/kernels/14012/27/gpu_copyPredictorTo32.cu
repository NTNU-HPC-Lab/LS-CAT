#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo32(int32_t * in, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{
int32_t * op = out + (index * theOutputPacketBytes) / 4;
op[stride * z] = (in + index * frameLength)[z];

}
}