#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void call_kALACSearch(int16_t * mCoefsU, int16_t * mCoefsV, int32_t kALACMaxCoefs)
{
int x = blockIdx.x;
int y = threadIdx.x;

int index = x * 16 * 16 + y * 16;
int32_t		k;
int32_t		den = 1 << DENSHIFT_DEFAULT;

mCoefsU[index + 0] = (AINIT * den) >> 4;
mCoefsU[index + 1] = (BINIT * den) >> 4;
mCoefsU[index + 2] = (CINIT * den) >> 4;

mCoefsV[index + 0] = (AINIT * den) >> 4;
mCoefsV[index + 1] = (BINIT * den) >> 4;
mCoefsV[index + 2] = (CINIT * den) >> 4;

for (k = 3; k < kALACMaxCoefs; k++)
{
mCoefsU[index + k] = 0;
mCoefsV[index + k] = 0;
}
}