#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_unmix20(int32_t * u, int32_t * v, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{

int32_t		l, r;
uint8_t * op = out + (index * theOutputPacketBytes) ;

if (mixres[index] != 0)
{
/* matrixed stereo */
l = (u + index * frameLength)[z] + (v + index * frameLength)[z] - ((mixres[index] * (v + index * frameLength)[z]) >> mixbits[index]);
r = l - (v + index * frameLength)[z];

l <<= 4;
r <<= 4;

op += 3 * z;
op += (stride - 1) * 3 * z;
op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
op += 3;

op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
}
else
{
/* Conventional separated stereo. */
int32_t		val;

val = (u + index * frameLength)[z] << 4;
op += 3 * z;
op += (stride - 1) * 3 * z;
op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
op += 3;

val = (v + index * frameLength)[z] << 4;
op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
}
}
}