#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo16(int32_t * in, int16_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
int block = blockIdx.x % 8;
int index = blockIdx.x / 8;
int z = threadIdx.x + block * blockDim.x;

if (z < numSamples[index])
{
int16_t * op = out + (index * theOutputPacketBytes) / 2;

op[z] = (int16_t)(in + index * frameLength)[z];
}
}