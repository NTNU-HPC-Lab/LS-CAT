#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMatrix(int *c, int *a, int *b){
int j = blockIdx.x*blockDim.x + threadIdx.x;
int i = blockIdx.y*blockDim.y + threadIdx.y;
*(c + blockDim.y*i + j) = *(a + blockDim.y*i + j) + *(b + blockDim.y*i + j);
}