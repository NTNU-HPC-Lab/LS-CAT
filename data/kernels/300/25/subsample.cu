#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
// iterators
int xx, yy;

// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;
int k = blockIdx.x % input_n;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
output = output + o*output_w*output_h;
input = input + i*input_w*input_h;

// Get the good mask for (k,i) (k out, i in)
float the_weight = weight[k];

// Initialize to the bias
float the_bias = bias[k];

// For all output pixels...
for(yy = yy_start; yy < yy_end; yy+=yy_step) {
for(xx = xx_start; xx < xx_end; xx+=xx_step) {
// Compute the mean of the input image...
float *ptr_input = input + yy*dH*input_w + xx*dW;
float *ptr_output = output + yy*output_w + xx;
float sum = 0;
int kx, ky;
for(ky = 0; ky < kH; ky++) {
for(kx = 0; kx < kW; kx++)
sum += ptr_input[kx];
ptr_input += input_w; // next input line
}
// Update output
*ptr_output = the_weight*sum + the_bias;
}
}
}