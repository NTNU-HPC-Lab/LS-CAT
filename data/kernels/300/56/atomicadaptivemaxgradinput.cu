#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
// iterators
int xx, yy;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
gradOutput = gradOutput + o*output_w*output_h;
gradInput = gradInput + i*input_w*input_h;
indices_x = indices_x + o*output_w*output_h;
indices_y = indices_y + o*output_w*output_h;

// compute gradInput
for(yy = yy_start; yy < yy_end; yy+=yy_step) {

int y_start = (int)floor(float(yy) / output_h * input_h);

for(xx = xx_start; xx < xx_end; xx+=xx_step) {

int x_start = (int)floor(float(xx) / output_w * input_w);

float *ptr_gradInput = gradInput + y_start*input_w + x_start;
float *ptr_gradOutput = gradOutput + yy*output_w + xx;
float *ptr_ind_x = indices_x + yy*output_w + xx;
float *ptr_ind_y = indices_y + yy*output_w + xx;
float z = *ptr_gradOutput;

int argmax_x = (*ptr_ind_x)-1;
int argmax_y = (*ptr_ind_y)-1;

// atomic add since different threads could update same variable
atomicAdd(&(ptr_gradInput[argmax_x + argmax_y*input_w]), z);
}
}
}