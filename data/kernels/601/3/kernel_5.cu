#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_5(int *new_data, int *data)
{
int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

if (_tid_ >= 10000000) return;

int idx_2 = (_tid_ / 2) % 500;

new_data[_tid_] = (data[_tid_] + idx_2) % 13377;
}