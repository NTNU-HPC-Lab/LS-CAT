#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREADS 511
#define ITERATIONS 100000

using namespace std;



__global__ void kernel_map(int *values, int *next_values)
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;

if (tid < NUM_THREADS)
{
next_values[tid] = values[tid] + 1;
}
}