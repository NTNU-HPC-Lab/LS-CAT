#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_stencil(float *new_data, float *data, float *param_a, float *param_b, float *param_c, float *param_wrk, float *param_bnd) {

int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

if (_tid_ >= 129 * 65 * 65) return;

int idx_0 =_tid_ / 65 / 65;
int idx_1 = (_tid_ / 65) % 65;
int idx_2 = (_tid_ / 1) % 65;

if (idx_0 - 1 < 0 || idx_0 + 1 >= 129) { new_data[_tid_] = 0.0; return; }
if (idx_1 - 1 < 0 || idx_2 + 1 >= 65) { new_data[_tid_] = 0.0; return; }
if (idx_1 - 1 < 0 || idx_2 + 1 >= 65) { new_data[_tid_] = 0.0; return; }

float v000 = data[(idx_0) * 65 * 65 + (idx_1) * 65 + (idx_2)];
float v100 = data[(idx_0 + 1) * 65 * 65 + (idx_1) * 65 + (idx_2)];
float v010 = data[(idx_0) * 65 * 65 + (idx_1 + 1) * 65 + (idx_2)];
float v001 = data[(idx_0) * 65 * 65 + (idx_1) * 65 + (idx_2 + 1)];
float v110 = data[(idx_0 + 1) * 65 * 65 + (idx_1 + 1) * 65 + (idx_2)];
float v120 = data[(idx_0 + 1) * 65 * 65 + (idx_1 - 1) * 65 + (idx_2)];
float v210 = data[(idx_0 - 1) * 65 * 65 + (idx_1 + 1) * 65 + (idx_2)];
float v220 = data[(idx_0 - 1) * 65 * 65 + (idx_1 - 1) * 65 + (idx_2)];
float v011 = data[(idx_0) * 65 * 65 + (idx_1 + 1) * 65 + (idx_2 + 1)];
float v021 = data[(idx_0) * 65 * 65 + (idx_1 - 1) * 65 + (idx_2 + 1)];
float v012 = data[(idx_0) * 65 * 65 + (idx_1 + 1) * 65 + (idx_2 - 1)];
float v022 = data[(idx_0) * 65 * 65 + (idx_1 - 1) * 65 + (idx_2 - 1)];
float v101 = data[(idx_0 + 1) * 65 * 65 + (idx_1) * 65 + (idx_2 + 1)];
float v201 = data[(idx_0 - 1) * 65 * 65 + (idx_1) * 65 + (idx_2 + 1)];
float v102 = data[(idx_0 + 1) * 65 * 65 + (idx_1) * 65 + (idx_2 - 1)];
float v202 = data[(idx_0 - 1) * 65 * 65 + (idx_1) * 65 + (idx_2 - 1)];
float v200 = data[(idx_0 - 1) * 65 * 65 + (idx_1) * 65 + (idx_2)];
float v020 = data[(idx_0) * 65 * 65 + (idx_1 - 1) * 65 + (idx_2)];
float v002 = data[(idx_0) * 65 * 65 + (idx_1) * 65 + (idx_2 - 1)];

new_data[_tid_] =
v000 + 0.8 * (((
param_a[65 * 65 * 4 * idx_0 + 65 * 4 * idx_1 + 4 * idx_2 + 0] * v100 +
param_a[65 * 65 * 4 * idx_0 + 65 * 4 * idx_1 + 4 * idx_2 + 1] * v010 +
param_a[65 * 65 * 4 * idx_0 + 65 * 4 * idx_1 + 4 * idx_2 + 2] * v001 +
param_b[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 0] *
(v110 - v120 - v210 + v220) +
param_b[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 1] *
(v011 - v021 - v012 + v022) +
param_b[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 2] *
(v101 - v201 - v102 + v202) +
param_c[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 0] * v200 +
param_c[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 1] * v020 +
param_c[65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2 + 2] * v002 +
param_wrk[65 * 65 * idx_0 + 65 * idx_1 + idx_2]) *
param_a[65 * 65 * 4 * idx_0 + 65 * 4 * idx_1 + 4 * idx_2 + 3] -
v000) * param_bnd[65 * 65 * idx_0 + 65 * idx_1 + idx_2]);
}