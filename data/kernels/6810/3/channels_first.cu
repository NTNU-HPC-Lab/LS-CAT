#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
// n (batch size), c (num of channels), y (height), x (width)
int n = blockIdx.x;
int y = blockIdx.y;
int x = blockIdx.z;

int ch_off = threadIdx.x;
float value;

int dimcyx = channels * height * width;
int dimyx = height * width;

int p_dimx = (width + 2 * pad_size);
int p_dimy = (height + 2 * pad_size);
int p_dimyxc = channels * p_dimy * p_dimx;
int p_dimxc = p_dimx * channels;

for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
value = input[n * dimcyx + c * dimyx + y * width + x];
rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = value;
}
}