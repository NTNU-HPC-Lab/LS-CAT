#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *a, int *b, int *c){
int my_x, my_y;
my_x = blockIdx.x*blockDim.x + threadIdx.x;
my_y = blockIdx.y*blockDim.y + threadIdx.y;
int local_c = 0;
for(int i = 0 ; i < size; i++)
local_c += a[my_x * size + i] * b[i * size + my_y];

c[my_x * size + my_y ] = local_c;

}