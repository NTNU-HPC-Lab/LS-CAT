#include "hip/hip_runtime.h"
#include "includes.h"
#define size 1024
#define block_size 32




__global__ void matrixMulOptimized(int* a, int* b, int* c)
{



__shared__ float a_share[32][32];
__shared__ float b_share[32][32];

int n = 1024;
int row = blockDim.y*blockIdx.y + threadIdx.y;
int col = blockDim.x*blockIdx.x + threadIdx.x;

int local_c = 0;
for (int i = 0; i < 32; ++i)
{
a_share[threadIdx.y][threadIdx.x] = a[row*n + i*blockDim.y + threadIdx.x];
b_share[threadIdx.y][threadIdx.x] = b[(i*blockDim.x + blockIdx.y)*n + col];

__syncthreads();
for (int k = 0; k < 32; ++k)
{
local_c += a_share[threadIdx.y][k]*b_share[k][threadIdx.x];
}
__syncthreads();

}

c[row*n + col] = local_c;
}