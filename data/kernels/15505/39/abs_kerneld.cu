#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void abs_kerneld(double *v, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = ::abs(v[x]);
}