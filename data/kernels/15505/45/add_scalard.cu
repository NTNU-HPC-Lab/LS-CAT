#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_scalard(double *v, double other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] += other;
}