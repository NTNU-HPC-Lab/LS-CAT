#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(float *v, int n, float e) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = ::pow(v[x], e);
}