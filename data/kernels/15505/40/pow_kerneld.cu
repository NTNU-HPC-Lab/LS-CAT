#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kerneld(double *v, int n, double e) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = ::pow(v[x], e);
}