#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clip_kerneld(double *v, int n, double limit) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = (v[x] > limit) ? limit : ((v[x] < -limit) ? -limit : v[x]);
}