#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sqrt_kernel(float *v, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = sqrt(v[x]);
}