#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divide_kernel(float *v, float *other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] /= other[x];
}