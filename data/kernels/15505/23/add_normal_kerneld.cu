#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_normal_kerneld(int seed, double *data, int n, double mean, double std) {
if (threadIdx.x != 0) return;
hiprandState state;
hiprand_init(seed, 0, 0, &state);
for (size_t i(0); i < n; ++i)
data[i] += hiprand_normal_double(&state) * std + mean;
}