#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times_scalard(double *v, double other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] *= other;
}