#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times_kernel(float *v, float *other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] *= other[x];
}