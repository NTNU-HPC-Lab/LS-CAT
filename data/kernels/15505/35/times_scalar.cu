#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times_scalar(float *v, float other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] *= other;
}