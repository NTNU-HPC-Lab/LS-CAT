#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_normal_kernel(int seed, float *data, int n, float mean, float std) {
if (threadIdx.x != 0) return;
hiprandState state;

hiprand_init(seed, 0, 0, &state);
for (size_t i(0); i < n; ++i)
data[i] += hiprand_normal(&state) * std + mean;
}