#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clip_kernel(float *v, int n, float limit) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = (v[x] > limit) ? limit : ((v[x] < -limit) ? -limit : v[x]);
}