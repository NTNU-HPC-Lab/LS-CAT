#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times_kerneld(double *v, double *other, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] *= other[x];
}