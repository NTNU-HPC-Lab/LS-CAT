#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void abs_kernel(float *v, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = ::abs(v[x]);
}