#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ void copy_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
// *out = *in;
for (size_t c(0); c < C; ++c)
out[c * slicesizeout] = in[c * slicesizein];
}
__device__ __forceinline__ void add_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
// *out = *in + *out;
for (size_t c(0); c < C; ++c)
out[c * slicesizeout] += in[c * slicesizein];
}
__device__ __forceinline__ int get_index(int X, int Y, int Z, int C, int x, int y, int z) {
return z * (C * X * Y) + y * X + x;
}
__global__ void shift_kernel(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
int y(x / X);
x = x % X;

int x_to(x + dx);
int y_to(y + dy);
// int x_to(x);
// int y_to(y);
// y = 0;
// y_to = 0;


if (x >= X || y >= Y || x_to >= X || y_to >= Y || x_to < 0 || y_to < 0)
return;
if (beta>0)
add_c(in + get_index(X, Y, 1, C, x, y, 0), out + get_index(X, Y, 1, C, x_to, y_to, 0), X * Y, X * Y, C);
else
copy_c(in + get_index(X, Y, 1, C, x, y, 0), out + get_index(X, Y, 1, C, x_to, y_to, 0), X * Y, X * Y, C);

}