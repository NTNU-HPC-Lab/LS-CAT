#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_kernel(float *v, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);
if (x >= n) return;

v[x] = exp(v[x]);
}