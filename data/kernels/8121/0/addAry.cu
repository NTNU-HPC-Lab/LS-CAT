#include "hip/hip_runtime.h"
#include "includes.h"
// CUDA-C includes



extern "C" void runCudaPart();




// Main cuda function

__global__ void addAry( int * ary1, int * ary2 )
{
int indx = threadIdx.x;
ary1[ indx ] += ary2[ indx ];
}