#include "hip/hip_runtime.h"
#include "includes.h"

#define Width 1920
#define Height 2520
#define iterations 100




__global__ void convolution_kernel(unsigned char* A, unsigned char* B)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int x = i-2*blockIdx.x-1;
int y = j-2*blockIdx.y-1;

__shared__ unsigned char As[32][32];

//Copy from global memory to shared memory

if (x<0) {
x=0;
} else if (x==Width) {
x=Width-1;
}
if (y<0) {
y=0;
} else if (y == Height) {
y = Height-1;
}
As[threadIdx.x][threadIdx.y] = A[Width*y + x];

__syncthreads();

// Computations

if (threadIdx.x!=0 && threadIdx.x!=31 && threadIdx.y!=0 && threadIdx.y!=31) {
B[Width*y + x] =     (As[threadIdx.x-1][threadIdx.y-1]  +
As[threadIdx.x  ][threadIdx.y-1] * 2 +
As[threadIdx.x+1][threadIdx.y-1]  +
As[threadIdx.x-1][threadIdx.y  ] *2 +
As[threadIdx.x  ][threadIdx.y  ] *4 +
As[threadIdx.x+1][threadIdx.y  ] * 2 +
As[threadIdx.x-1][threadIdx.y+1] * 1 +
As[threadIdx.x  ][threadIdx.y+1] * 2 +
As[threadIdx.x+1][threadIdx.y+1] * 1)/16;
}
}