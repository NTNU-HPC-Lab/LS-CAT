#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_4(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if(i<n_vertices)
{
int indexUsed = 0, indexOffset = 0;
int iStart = 0, iEnd = 0;
int k = 0;

if(i > 0)
{
k = d_sizeAdj[i-1];
indexOffset = d_LCMSize[i-1];
}

iEnd = d_sizeAdj[i];

for(int j = indexOffset; j<iEnd; j++)
{
d_lcmMatrix[j] = 0;
}

__syncthreads();

for(int j = 0; j < n_vertices; j++) {
if(i==j)
continue;
iStart = k;
int jStart = 0, jEnd = 0;

if(j > 0)
jStart = d_sizeAdj[j-1];
jEnd = d_sizeAdj[j];

int compVec = 0;

while (iStart < iEnd && jStart < jEnd)
{
if(d_adjList[iStart] < d_adjList[jStart])
iStart++;
else if (d_adjList[jStart] < d_adjList[iStart])
jStart++;
else // if arr1[i] == arr2[j]
{
jStart++;
iStart++;
compVec++;
}
}

if (compVec > 0)
{
atomicAdd((int*)&d_lcmMatrix[indexUsed + indexOffset], compVec);
// d_lcmMatrix[indexUsed + indexOffset] = compVec;
indexUsed++;
}
// __syncthreads();
}

// __syncthreads();
// d_LCMSize[i] = indexUsed;
// __syncthreads();

}

}