#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n) {

//
int vertex = blockIdx.x;
int vcomp = threadIdx.x;
bool equal;

//
__shared__ int cval;

//
if(vcomp == 0)
cval = 0;
__syncthreads();

//
if(vertex < n && vcomp < n)
for(int i = vcomp; i < n; i += blockDim.x) {

if(vertex == i) {
atomicAdd(&cval, 1);
continue;
}

if((lcmsizes[vertex+1] - lcmsizes[vertex]) != (lcmsizes[i+1] - lcmsizes[i]))
continue;

equal = false;

for(int j = 0; j < lcmsizes[vertex+1] - lcmsizes[vertex]; j++) {

if(lcm[lcmsizes[vertex] + j] == lcm[lcmsizes[i] + j])
equal = true;

else {
equal = false;
break;
}
}

if(equal)
atomicAdd(&cval, 1);
}

__syncthreads();
if(vertex < n && vcomp == 0 && cval > 0) {
atomicAdd(&hist[cval], 1);
//printf("\nv%d: %d\n", vertex, cval);
}
}