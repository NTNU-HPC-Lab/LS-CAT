#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_1_HIST(int* lcm, int* hist, int n) {

//
int vertex = blockIdx.x;
int vcomp = threadIdx.x;
bool equal;

//
__shared__ int cval;

//
if(vcomp == 0)
cval = 0;
__syncthreads();

//
if(vertex < n && vcomp < n)
for(int i = vcomp; i < n; i += blockDim.x) {

if(vertex == i) {
atomicAdd(&cval, 1);
continue;
}

equal = false;

for(int j = 0; j < n; j++) {

if(lcm[vertex*n + j] == lcm[i*n + j])
equal = true;

else {
equal = false;
break;
}
}

if(equal)
atomicAdd(&cval, 1);
}

__syncthreads();
if(vertex < n && vcomp == 0 && cval > 0) {
atomicAdd(&hist[cval], 1);
//printf("\nv%d: %d\n", vertex, cval);
}
}