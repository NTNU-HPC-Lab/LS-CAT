#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_4_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if(i<n_vertices)
{
int indexUsed = 0;
int iStart = 0, iEnd = 0;
int k = 0;

if(i > 0)
{
k = d_sizeAdj[i-1];
}

iEnd = d_sizeAdj[i];

__syncthreads();

for(int j = 0; j < n_vertices; j++) {
if(i==j)
continue;
iStart = k;
int jStart = 0, jEnd = 0;

if(j > 0)
jStart = d_sizeAdj[j-1];
jEnd = d_sizeAdj[j];

int compVec = 0;

while (iStart < iEnd && jStart < jEnd)
{
if(d_adjList[iStart] < d_adjList[jStart])
iStart++;
else if (d_adjList[jStart] < d_adjList[iStart])
jStart++;
else // if arr1[i] == arr2[j]
{
jStart++;
iStart++;
compVec++;
break;
}
}

if (compVec > 0)
{
indexUsed++;
}
}

__syncthreads();
d_LCMSize[i] = indexUsed;
// __syncthreads();

}

}