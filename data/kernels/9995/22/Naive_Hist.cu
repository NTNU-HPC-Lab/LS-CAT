#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices) {

//each block compares the same row to all others row2
int row = blockIdx.x;
int row2 = threadIdx.x;
bool equal;

//shared count for whole block/same vertice
__shared__ int count;

//one thread sets count to zero and syncsthreads.
if(row2 == 0)
count = 0;
__syncthreads();

//checks equality to other vertices
if(row < n_vertices && row2 < n_vertices)
for(int i = row2; i < n_vertices; i += blockDim.x) {

//checks equality of vertices lcm
equal = false;
for(int j = 0; j < n_vertices; j++) {

if(d_result[row*n_vertices +j] == d_result[i*n_vertices + j])
equal = true;
else {
equal = false;
break;
}
}

//adds to count if vertices are equal
if(equal)
atomicAdd(&count, 1);
}

//syncsthreads so count is done and increments hist[count]
__syncthreads();
if(row < n_vertices && row2 == 0 && count > 0)
atomicAdd(&d_hist[count], 1);
}