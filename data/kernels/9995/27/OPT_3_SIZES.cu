#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n) {

int vertex = blockIdx.x;
int vcomp = threadIdx.x;
int cval;

if(vertex < n && vcomp < n)
for(int i = vcomp; i < n; i += blockDim.x) {

//skips to next vertex
if(vertex == i) {
continue;
}

//resets count
cval = 0;

//for loop that goes through vertex neighbors
for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

//loop compares to other vertex i/vcomp
for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

++cval;
break;
}
}

if(cval > 0) {
atomicAdd(&lcmsizes[vertex + 1], 1);
break;
}
}
}
}