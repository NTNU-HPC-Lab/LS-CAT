#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OPT_4_HIST(int *d_lcmMatrix, int *d_LCMSize, int *d_histogram, int n_vertices)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
int count = 0, countMax = -1;

if(i<n_vertices)
{
int iStart = 0;
if(i>0)
iStart = d_LCMSize[i - 1]; //Offset
count = 0;
int iSize = d_LCMSize[i] - iStart;

for(int j = 0; j < n_vertices; j++) {
int jStart = 0;
if(j>0)
jStart = d_LCMSize[j - 1]; //Offset

int jSize = d_LCMSize[j] - jStart;
if(iSize != jSize)
continue;

int eq = 1;
for(int k = 0; k < iSize; k++)
{
if(d_lcmMatrix[iStart + k] != d_lcmMatrix[jStart + k])
{
eq = 0;
break;
}
}
if(eq == 1)
{
count++;
}
}

if(countMax < count)
countMax = count;
atomicAdd((int*)&d_histogram[count], 1);
// d_histogram[count]++;
}
}