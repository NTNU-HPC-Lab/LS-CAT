#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constructCircuitGraphVertex(unsigned int * C,unsigned int * offset,unsigned int ecount, unsigned int * cv, unsigned int cvCount){
unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
if(tid < ecount){
if(C[tid]!=0){
cv[offset[tid]]=tid;
}
}
}