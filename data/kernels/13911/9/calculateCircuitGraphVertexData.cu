#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateCircuitGraphVertexData( unsigned int * D,unsigned int * C,unsigned int ecount){

unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
if( tid <ecount)
{
unsigned int c=D[tid];
atomicExch(C+c,1);
}
}