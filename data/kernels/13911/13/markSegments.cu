#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void markSegments(	unsigned short * d_mark, unsigned int 	circuitGraphEdgeCount, unsigned int * 	d_cg_edge_start, unsigned int *	d_cedgeCount, unsigned int 	circuitVertexSize){

unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
if(tid<circuitVertexSize){
d_mark[ d_cg_edge_start[tid]]=d_cedgeCount[tid];
}

}