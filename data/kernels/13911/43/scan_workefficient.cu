#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_workefficient(float *g_odata, float *g_idata, int n)
{
// Dynamically allocated shared memory for scan kernels
extern  __shared__  float temp[];

int thid = threadIdx.x;

int offset = 1;

// Cache the computational window in shared memory
temp[2*thid]   = g_idata[2*thid];
temp[2*thid+1] = g_idata[2*thid+1];

// build the sum in place up the tree
for (int d = n>>1; d > 0; d >>= 1)
{
__syncthreads();

if (thid < d)
{
int ai = offset*(2*thid+1)-1;
int bi = offset*(2*thid+2)-1;

temp[bi] += temp[ai];
}

offset *= 2;
}

// scan back down the tree

// clear the last element
if (thid == 0)
{
temp[n - 1] = 0;
}

// traverse down the tree building the scan in place
for (int d = 1; d < n; d *= 2)
{
offset >>= 1;
__syncthreads();

if (thid < d)
{
int ai = offset*(2*thid+1)-1;
int bi = offset*(2*thid+2)-1;

float t   = temp[ai];
temp[ai]  = temp[bi];
temp[bi] += t;
}
}

__syncthreads();

// write results to global memory
g_odata[2*thid]   = temp[2*thid];
g_odata[2*thid+1] = temp[2*thid+1];
}