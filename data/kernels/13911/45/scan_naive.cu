#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_naive(float *g_odata, float *g_idata, int n)
{
// Dynamically allocated shared memory for scan kernels
extern  __shared__  float temp[];

int thid = threadIdx.x;

int pout = 0;
int pin = 1;

// Cache the computational window in shared memory
temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;

for (int offset = 1; offset < n; offset *= 2)
{
pout = 1 - pout;
pin  = 1 - pout;
__syncthreads();

temp[pout*n+thid] = temp[pin*n+thid];

if (thid >= offset)
temp[pout*n+thid] += temp[pin*n+thid - offset];
}

__syncthreads();

g_odata[thid] = temp[pout*n+thid];
}