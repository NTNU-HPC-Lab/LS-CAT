#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
__shared__ float shmem[DP_BLOCKSIZE];

uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
shmem[threadIdx.x] = 0;
if (eidx < gridDim.x * DP_BLOCKSIZE) {
for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
shmem[threadIdx.x] += a[eidx] * b[eidx];
}
}
__syncthreads();
if (threadIdx.x < 256) {
shmem[threadIdx.x] += shmem[threadIdx.x + 256];
}
__syncthreads();
if (threadIdx.x < 128) {
shmem[threadIdx.x] += shmem[threadIdx.x + 128];
}
__syncthreads();
if (threadIdx.x < 64) {
shmem[threadIdx.x] += shmem[threadIdx.x + 64];
}
__syncthreads();
if (threadIdx.x < 32) {
volatile float* mysh = &shmem[threadIdx.x];
*mysh += mysh[32];
*mysh += mysh[16];
*mysh += mysh[8];
*mysh += mysh[4];
*mysh += mysh[2];
*mysh += mysh[1];
if (threadIdx.x == 0) {
target[blockIdx.x] = *mysh;
}
}
}