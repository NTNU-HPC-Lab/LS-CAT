#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDot_m1T_m2(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows, const int m1_columns, const int m2_columns ){
/*  Increments the output matrix with the product of two matrices: m1 transposed and m2.
Inputs:
m1: array, left matrix of size m1_rows x m1_columns (m1 transposed will be of size m1_columns x m1_rows)
m2: array, right matrix of size m1_rows x m2_columns
output: array, the results of the computation are to be stored here:
m1 * m2, product of two arrays m1 and m2, a matrix of size m1_columns x m2_columns
m1_rows: int, number of rows in the left matrix m1
m1_columns: int, number of columns in the left matrix m1
m2_rows: int, number of rows in the left matrix m2
*/

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
int r = (int)i / m2_columns;
int c = i % m2_columns;
int id_T;
float t_output = 0.0;

for( int k = 0; k < m1_rows; ++k ) {
id_T = k * m1_columns + r;
t_output += m1[ id_T ] * m2[ k * m2_columns + c ];
}

output[i] += t_output;
}
}