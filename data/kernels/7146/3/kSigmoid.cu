#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSigmoid(const int nThreads, float const *input, float *output){
/*  Computes the value of the sigmoid function f(x) = 1/(1 + e^-x).
Inputs:
input: array
output: array, the results of the computation are to be stored here
*/

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = 1.0 / (1.0 + std::exp(-input[i]));
}
}