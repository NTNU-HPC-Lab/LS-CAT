#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMartixByMatrixElementwise(const int nThreads, const float *m1, const float *m2, float *output) {
/*  Computes the product of two arrays (elementwise multiplication).
Inputs:
m1: array
m2: array
output: array,the results of the multiplication are to be stored here
*/
for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = m1[i] * m2[i];
}
}