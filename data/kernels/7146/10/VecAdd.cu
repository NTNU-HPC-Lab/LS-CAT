#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const int* A, const int* B, int* C, int N) {

// Index holen
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < N)
C[i] = A[i] + B[i];

}