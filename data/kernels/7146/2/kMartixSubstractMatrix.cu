#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMartixSubstractMatrix(const int nThreads, const float *m1, const float *m2, float *output) {
/*  Computes the (elementwise) difference between two arrays
Inputs:
m1: array
m2: array
output: array,the results of the computation are to be stored here
*/

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = m1[i] - m2[i];
}
}