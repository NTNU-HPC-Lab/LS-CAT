#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
/*  Computes the product of two matrices: m1 x m2.
Inputs:
m1: array, left matrix of size m1_rows x m1_columns
m2: array, right matrix of size m1_columns x m2_columns (the number of rows in the right matrix
must be equal to the number of the columns in the left one)
output: array, the results of the computation are to be stored here:
m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_columns
m1_rows: int, number of rows in the left matrix m1
m1_columns: int, number of columns in the left matrix m1
m2_columns: int, number of columns in the right matrix m2
*/

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
int r = (int)i / m2_columns;
int c = i % m2_columns;
float t_output = 0.f;

for( int k = 0; k < m1_columns; ++k ) {
t_output += m1[ r * m1_columns + k ] * m2[ k * m2_columns + c ];
}

output[i] = t_output;
}
}