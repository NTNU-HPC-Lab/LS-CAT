#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSigmoid_d(const int nThreads, float const *input, float *output) {
/*  Computes the value of the sigmoid function derivative f'(x) = f(x)(1 - f(x)),
where f(x) is sigmoid function.
Inputs:
input: array
output: array, the results of the computation are to be stored here:
x(1 - x) for every element of the input matrix m1.
*/

for (int i = blockIdx.x * blockDim.x + threadIdx.x;
i < nThreads;
i += blockDim.x * gridDim.x)
{
output[i] = input[i] * (1 - input[i]);
}
}