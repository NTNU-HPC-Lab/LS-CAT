#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_arrays_kernel(int64_t* __restrict a, const int64_t* __restrict b, const int64_t array_count) {
for (int i = threadIdx.x + (blockIdx.x * blockDim.x); i < array_count; i += blockDim.x * gridDim.x) {
a[i] += b[i];
}
}