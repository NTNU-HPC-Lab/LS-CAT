#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n) {
return;
}

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[size];
int *s_c = &s_b[size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

if (s_a[tid] <= s_b[tid])
{
s_c[tid] = s_a[tid];
}
else
{
s_c[tid] = s_b[tid];
}
c[tid] = s_c[tid];
}