#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid>= n)
{
return;
}

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[n];
int *s_res = &s_b[n];

__shared__ int s_k, s_p;

s_k = k;
s_p = p;

s_a[tid] = a[tid];
s_b[tid] = b[tid];
s_res[tid] = res[tid];

s_res[tid] = s_a[tid] + s_b[tid];
if (s_res[tid] > s_k)
{
s_res[tid] = s_p;
}

res[tid] = s_res[tid];
}