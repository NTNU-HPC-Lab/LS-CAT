#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int * a, int *b, int *res, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

__shared__ int s_a[size], s_b[size], s_res[size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
if (s_res[tid] < 0)
{
s_res[tid] = 0;
}
res[tid] = s_res[tid];
}