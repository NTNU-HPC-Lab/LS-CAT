#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *res, int x, int n) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid < n) {
res[tid] = a[tid] - (b[tid] * x);
}
}