#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateMatrixFormulaSharedStatic(int *a, int *b, int *res, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

__shared__ int s_a[size * size], s_b[size * size], s_res[size * size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
res[tid] = s_res[tid];
}