#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *niz, int *res, int k, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n) {
return;
}

if (niz[tid] == k) {
atomicAdd(res, 1);
}
}