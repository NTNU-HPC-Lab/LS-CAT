#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n) {
return;
}

__shared__ int s_a[size], s_b[size], s_c[size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

if (s_a[tid] >= s_b[tid])
{
s_c[tid] = s_a[tid];
}
else
{
s_c[tid] = s_b[tid];
}
c[tid] = s_c[tid];
}