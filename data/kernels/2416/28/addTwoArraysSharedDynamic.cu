#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addTwoArraysSharedDynamic(int *v1, int *v2, int *r, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

extern __shared__ int arrays[];
int *s_v1 = arrays;
int *s_v2 = &s_v1[n];
int *s_r = &s_v2[n];

s_v1[tid] = v1[tid];

s_v2[tid] = v2[tid];

s_r[tid] = s_v1[tid] + s_v2[tid];
r[tid] = s_r[tid];
}