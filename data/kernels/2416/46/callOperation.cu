#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int x, int *res, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid < n) {
res[tid] = ((a[tid] * x) + b[tid]);
}
}