#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int * a, int *b, int *res, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}
int tid = tidx * n + tidy;

__shared__ int s_a[size * size], s_b[size * size], s_res[size * size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
if (s_res[tid] < 0)
{
s_res[tid] = 0;
}
res[tid] = s_res[tid];
}