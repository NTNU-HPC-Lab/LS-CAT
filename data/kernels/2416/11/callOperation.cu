#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *res, int x, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

res[tid] = a[tid] * x;
}