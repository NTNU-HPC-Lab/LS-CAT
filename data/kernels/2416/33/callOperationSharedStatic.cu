#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}

int tid = tidx * n + tidy;

__shared__ int s_a[size * size], s_b[size * size], s_c[size * size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

if (s_a[tid] <= s_b[tid])
{
s_c[tid] = s_a[tid];
}
else
{
s_c[tid] = s_b[tid];
}

c[tid] = s_c[tid];
}