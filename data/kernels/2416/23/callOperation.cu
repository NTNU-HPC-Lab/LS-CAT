#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *c, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}

int tid = tidx * n + tidy;

if (a[tid] >= b[tid])
{
c[tid] = a[tid];
}
else
{
c[tid] = b[tid];
}
}