#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *res, int x, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

extern __shared__ int data[];

int *s_a = data;
int *s_res = &s_a[size * size];

__shared__ int s_x;

s_x = x;
s_a[tid] = a[tid];

s_res[tid] = s_a[tid] * s_x;

res[tid] = s_res[tid];
}