#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int * a, int *b, int *res, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

res[tid] = a[tid] - b[tid];
if (res[tid] < 0)
{
res[tid] = 0;
}
}