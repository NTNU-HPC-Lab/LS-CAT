#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int x, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

extern __shared__ int arrays[];
__shared__ int s_x;

int *s_a = arrays;
int *s_b = &s_a[n];
int *s_res = &s_b[n];

s_x = x;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - (s_b[tid] * s_x);
res[tid] = s_res[tid];
}