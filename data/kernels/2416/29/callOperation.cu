#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n) {
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n) {
return;
}

res[tid] = a[tid] - b[tid];
if (res[tid] < k) {
res[tid] = p;
}
}