#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addTwoArrays(int *v1, int *v2, int *r, int n)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid >= n) {
return;
}

r[tid] = v1[tid] + v2[tid];
}