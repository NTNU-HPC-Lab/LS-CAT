#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int idy = blockDim.y * blockIdx.y + threadIdx.y;

if (idx >= n || idy >= n)
{
return;
}

int tid = idx * n + idy;

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[size * size];
int *s_res = &s_b[size * size];

__shared__ int s_k, s_p;

s_k = k;
s_p = p;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] + s_b[tid];

if (s_res[tid] > s_k)
{
s_res[tid] = s_p;
}

res[tid] = s_res[tid];
}