#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *result, int k, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}

int tid = tidx * n + tidy;

if (a[tid] == k)
{
atomicAdd(result, 1);
}
}