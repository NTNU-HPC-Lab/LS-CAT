#include "hip/hip_runtime.h"
#include "includes.h"


#define threads 32
#define size 5

using namespace std;




__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[size * size];
int *s_res = &s_b[size * size];

__shared__ int s_p, s_k;

s_k = k;
s_p = p;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
if (s_res[tid] < s_k) {
s_res[tid] = s_p;
}

res[tid] = s_res[tid];
}