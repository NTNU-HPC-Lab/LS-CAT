#include "hip/hip_runtime.h"
#include "includes.h"


#define threads 32
#define size 5

using namespace std;




__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

__shared__ int s_a[size * size], s_b[size * size], s_res[size * size], s_p, s_k;

s_k = k;
s_p = p;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
if (s_res[tid] < s_k) {
s_res[tid] = s_p;
}

res[tid] = s_res[tid];
}