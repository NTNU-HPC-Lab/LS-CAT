#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *c, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n) {
return;
}

if (a[tid] <= b[tid])
{
c[tid] = a[tid];
}
else
{
c[tid] = b[tid];
}
}