#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int * a, int *b, int *res, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}
int tid = tidx * n + tidy;

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[size * size];
int *s_res = &s_b[size * size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - s_b[tid];
if (s_res[tid] < 0)
{
s_res[tid] = 0;
}
res[tid] = s_res[tid];
}