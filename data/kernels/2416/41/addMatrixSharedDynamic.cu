#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMatrixSharedDynamic(int *a, int *b, int *res, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}

int tid = tidx * n + tidy;

extern __shared__ int arrays[];

int *s_a = arrays;
int *s_b = &arrays[size * size];
int *s_res = &s_b[size*size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] + s_b[tid];
res[tid] = s_res[tid];
}