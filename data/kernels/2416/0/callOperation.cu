#include "hip/hip_runtime.h"
#include "includes.h"


#define threads 32
#define size 5

using namespace std;




__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

res[tid] = a[tid] - b[tid];
if (res[tid] < k) {
res[tid] = p;
}
}