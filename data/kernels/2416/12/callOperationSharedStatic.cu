#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *res, int x, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;

__shared__ int s_a[size * size], s_res[size * size], s_x;

s_x = x;
s_a[tid] = a[tid];

s_res[tid] = s_a[tid] * s_x;

res[tid] = s_res[tid];
}