#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int idy = blockDim.y * blockIdx.y + threadIdx.y;

if (idx >= n || idy >= n)
{
return;
}

int tid = idx * n + idy;

__shared__ int s_a[size * size], s_b[size * size], s_res[size * size], s_k, s_p;

s_k = k;
s_p = p;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] + s_b[tid];

if (s_res[tid] > s_k)
{
s_res[tid] = s_p;
}

res[tid] = s_res[tid];
}