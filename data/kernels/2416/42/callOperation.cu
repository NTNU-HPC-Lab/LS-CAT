#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n)
{
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int idy = blockDim.y * blockIdx.y + threadIdx.y;

if (idx >= n || idy >= n) {
return;
}

int tid = idx * n + idy;

res[tid] = a[tid] + b[tid];

if (res[tid] > k) {
res[tid] = p;
}
}