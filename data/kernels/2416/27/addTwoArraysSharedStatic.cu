#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addTwoArraysSharedStatic(int *v1, int *v2, int *r, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

__shared__ int s_v1[SIZE], s_v2[SIZE], s_r[SIZE];

s_v1[tid] = v1[tid];

s_v2[tid] = v2[tid];

s_r[tid] = s_v1[tid] + s_v2[tid];
r[tid] = s_r[tid];
}