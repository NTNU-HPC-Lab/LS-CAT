#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n)
{
return;
}

int tid = tidx * n + tidy;

extern __shared__ int data[];

int *s_a = data;
int *s_b = &s_a[size * size];
int *s_c = &s_b[size * size];

s_a[tid] = a[tid];
s_b[tid] = b[tid];

if (s_a[tid] >= s_b[tid])
{
s_c[tid] = s_a[tid];
}
else
{
s_c[tid] = s_b[tid];
}

c[tid] = s_c[tid];
}