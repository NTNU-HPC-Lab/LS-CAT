#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMatrix(int *a, int *b, int *res, int n)
{
int tidx = blockDim.x * blockIdx.x + threadIdx.x;
int tidy = blockDim.y * blockIdx.y + threadIdx.y;

if (tidx >= n || tidy >= n) {
return;
}

int tid = tidx * n + tidy;
res[tid] = a[tid] + b[tid];
}