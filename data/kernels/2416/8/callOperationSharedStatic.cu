#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *b, int *res, int x, int n)
{
int tid = blockDim.x * blockIdx.x + threadIdx.x;

if (tid >= n)
{
return;
}

__shared__ int s_a[size], s_b[size], s_res[size];
__shared__ int s_x;

s_x = x;
s_a[tid] = a[tid];
s_b[tid] = b[tid];

s_res[tid] = s_a[tid] - (s_b[tid] * s_x);
res[tid] = s_res[tid];
}