#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define THREADS_PER_BLOCK 32
#define NUM_BLOCKS 32

typedef double HighlyPrecise;

const int GENOME_LENGTH = 14;
const int GENE_MAX = 1;

const float MUTATION_FACTOR = 0.2;
const float CROSSOVER_RATE = 0.6;

const int NUM_EPOCHS = 1000;

struct Chromosome {
HighlyPrecise genes[GENOME_LENGTH];
HighlyPrecise fitnessValue;
};


__global__ void setupRandomStream(unsigned int seed, hiprandState* states) {
int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
hiprand_init(seed, threadIndex, 0, &states[threadIndex]);
}