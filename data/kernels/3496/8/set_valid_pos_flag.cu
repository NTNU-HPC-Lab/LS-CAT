#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
const int32_t step = blockDim.x * gridDim.x;
for (int32_t i = start; i < entry_count; i += step) {
if (count_buff[i]) {
pos_buff[i] = VALID_POS_FLAG;
}
}
}