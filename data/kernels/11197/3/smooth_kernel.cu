#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= n) return;

int j = id % w;
id /= w;
int i = id % h;
id /= h;
int k = id % c;
id /= c;
int b = id;

int w_offset = -(size/2.);
int h_offset = -(size/2.);

int out_index = j + w*(i + h*(k + c*b));
int l, m;
for(l = 0; l < size; ++l){
for(m = 0; m < size; ++m){
int cur_h = h_offset + i + l;
int cur_w = w_offset + j + m;
int index = cur_w + w*(cur_h + h*(k + b*c));
int valid = (cur_h >= 0 && cur_h < h &&
cur_w >= 0 && cur_w < w);
delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
}
}
}