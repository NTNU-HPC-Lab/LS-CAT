#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "col2im_gpu_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int n = 1;
const float *data_col = NULL;
hipMalloc(&data_col, XSIZE*YSIZE);
const int height = 1;
const int width = 1;
const int ksize = 1;
const int pad = 1;
const int stride = 1;
const int height_col = 1;
const int width_col = 1;
float *data_im = NULL;
hipMalloc(&data_im, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
col2im_gpu_kernel<<<gridBlock,threadBlock>>>(n,data_col,height,width,ksize,pad,stride,height_col,width_col,data_im);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
col2im_gpu_kernel<<<gridBlock,threadBlock>>>(n,data_col,height,width,ksize,pad,stride,height_col,width_col,data_im);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
col2im_gpu_kernel<<<gridBlock,threadBlock>>>(n,data_col,height,width,ksize,pad,stride,height_col,width_col,data_im);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}