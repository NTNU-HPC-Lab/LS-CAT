#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
const int threads = BLOCK;
__shared__ float local[threads];

int id = threadIdx.x;
local[id] = 0;

int filter = blockIdx.x;

int i, j;
for(j = 0; j < batch; ++j){
for(i = 0; i < spatial; i += threads){
int index = j*spatial*filters + filter*spatial + i + id;

local[id] += (i+id < spatial) ? pow((x[index] - mean[filter]), 2) : 0;
}
}

__syncthreads();

if(id == 0){
variance[filter] = 0;
for(i = 0; i < threads; ++i){
variance[filter] += local[i];
}
variance[filter] /= (spatial * batch - 1);
}
}