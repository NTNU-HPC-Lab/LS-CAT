#include "hip/hip_runtime.h"
#include "includes.h"
// VectorAdd.cu




#define N 10 // size of vectors

#define B 1 // blocks in the grid

#define T 10 // threads in a block


__global__ void add (int *a,int *b, int *c) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < N) {
c[tid] = a[tid]+b[tid];
}
}