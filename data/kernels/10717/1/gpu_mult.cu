#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mult(int *a,int *b, int *c, int N) {

int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if( col < N && row < N) {
for(int i = 0; i < N; i++) {
sum += a[row * N + i] * b[i * N + col];
}
c[row * N + col] = sum;
}
}