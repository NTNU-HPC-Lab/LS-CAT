#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_map_backward_kernel_w(const float *dout, const float *weight, const float *g, float *dw, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int z = blockIdx.z;

if (x < width && y < height && z < height+width-1) {

for (int batch = 0; batch < num; ++batch) {
for (int plane = 0; plane < chn; ++plane) {
float _dout = dout[(batch * chn + plane) * sp + y*width + x];

if (z < width) {
int i = z;
float _g = g[(batch * chn + plane) * sp + y*width + i];
dw[(batch * len + i) * sp + y*width + x] += _dout * _g;
} else {
int i = z - width;
int j = i<y ? i : i+1;

float _g = g[(batch * chn + plane) * sp + j*width + x];
dw[(batch * len + width + i) * sp + y*width + x] += _dout * _g;
}
}
}
}
}