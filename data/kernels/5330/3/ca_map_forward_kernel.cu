#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_map_forward_kernel(const float *weight, const float *g, float *out, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int plane = blockIdx.z;

if (x < width && y < height && plane < chn) {
for (int batch = 0; batch < num; ++batch) {

for (int i = 0; i < width; ++i) {
float _g = g[(batch * chn + plane) * sp + y*width + i];
float _w = weight[(batch * len + i) * sp + y*width + x];
out[(batch * chn + plane) * sp + y*width + x] += _g * _w;
}
for (int i = 0; i < height; ++i) {
if (i == y) continue;

int j = i<y ? i : i-1;

float _g = g[(batch * chn + plane) * sp + i*width + x];
float _w = weight[(batch * len + width + j) * sp + y*width + x];
out[(batch * chn + plane) * sp + y*width + x] += _g * _w;
}
}
}

}