#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_forward_kernel(const float *t, const float *f, float *weight, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int z = blockIdx.z;

if (x < width && y < height && z < height+width-1) {
for (int batch = 0; batch < num; ++batch) {
for (int plane = 0; plane < chn; ++plane) {
float _t = t[(batch * chn + plane) * sp + y*width + x];

if (z < width) {
int i = z;
float _f = f[(batch * chn + plane) * sp + y*width + i];
weight[(batch * len + i) * sp + y*width + x] += _t*_f;
} else {
int i = z - width;
int j = i<y ? i : i+1;

float _f = f[(batch * chn + plane) * sp + j*width + x];
weight[(batch * len + width + i) * sp + y*width + x] += _t*_f;
}
}
}
}
}