#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_kernel(const float *x, const float *mean, const float *var, const float *weight, const float *bias, float *y, float *z, float eps, int N, int C, int S) {
int plane = blockIdx.x;

float _mean = mean[plane];
float _var = var[plane];
float invStd = 0;
if (_var != 0.f || eps != 0.f) {
invStd = 1 / sqrt(_var + eps);
}

float gamma = weight != 0 ? abs(weight[plane]) + eps : 1.f;
float beta = bias != 0 ? bias[plane] : 0.f;
for (int batch = 0; batch < N; ++batch) {
for (int n = threadIdx.x; n < S; n += blockDim.x) {
float _x = x[(batch * C + plane) * S + n];
float _y = (_x - _mean) * invStd;
float _z = _y * gamma + beta;

y[(batch * C + plane) * S + n] = _y;
z[(batch * C + plane) * S + n] = _z;
}
}
}