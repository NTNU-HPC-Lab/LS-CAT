#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_map_backward_kernel_g(const float *dout, const float *weight, const float *g, float *dg, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int plane = blockIdx.z;

if (x < width && y < height && plane < chn) {

for (int batch = 0; batch < num; ++batch) {
for (int i = 0; i < width; ++i) {
float _dout = dout[(batch * chn + plane) * sp + y*width + i];
float _w = weight[(batch * len + x) * sp + y*width + i];
dg[(batch * chn + plane) * sp + y*width + x] += _dout * _w;
}

for (int i = 0; i < height; ++i) {
if (i == y) continue;
int j = i>y ? y : y-1;

float _dout = dout[(batch * chn + plane) * sp + i*width + x];
float _w = weight[(batch * len + width + j) * sp + i*width + x];
dg[(batch * chn + plane) * sp + y*width + x] += _dout * _w;
}
}
}
}