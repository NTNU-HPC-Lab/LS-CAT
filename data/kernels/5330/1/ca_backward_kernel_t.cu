#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_backward_kernel_t(const float *dw, const float *t, const float *f, float *dt, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int plane = blockIdx.z;

if (x < width && y < height && plane < chn) {
for (int batch = 0; batch < num; ++batch) {

for (int i = 0; i < width; ++i) {
float _dw = dw[(batch * len + i) * sp + y*width + x];
float _f = f[(batch * chn + plane) * sp + y*width + i];
dt[(batch * chn + plane) * sp + y*width + x] += _dw * _f;
}
for (int i = 0; i < height; ++i)  {
if (i == y) continue;
int j = i<y ? i : i-1;

float _dw = dw[(batch * len + width + j) * sp + y*width + x];
float _f = f[(batch * chn + plane) * sp + i*width + x];
dt[(batch * chn + plane) * sp + y*width + x] += _dw * _f;
}
}

}
}