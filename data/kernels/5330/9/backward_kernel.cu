#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_kernel(const float *dz, const float *z, const float *var, const float *weight, const float *bias, const float *edz, const float *eydz, float *dx, float *dweight, float *dbias, float eps, int N, int C, int S) {
int plane = blockIdx.x;
float _edz = edz[plane];
float _eydz = eydz[plane];

float gamma = weight != 0 ? abs(weight[plane]) + eps : 1.f;
float beta = bias != 0 ? bias[plane] : 0.f;

if (dx != 0) {
float _var = var[plane];
float invStd = 0;
if (_var != 0.f || eps != 0.f) {
invStd = 1 / sqrt(_var + eps);
}

float mul = gamma * invStd;

for (int batch = 0; batch < N; ++batch) {
for (int n = threadIdx.x; n < S; n += blockDim.x) {
float _dz = dz[(batch * C + plane) * S + n];
float _y = (z[(batch * C + plane) * S + n] - beta) / gamma;
dx[(batch * C + plane) * S + n] = (_dz - _edz - _y * _eydz) * mul;
}
}
}

if (dweight != 0 || dbias != 0) {
float norm = N * S;

if (dweight != 0) {
if (threadIdx.x == 0) {
if (weight[plane] > 0)
dweight[plane] += _eydz * norm;
else if (weight[plane] < 0)
dweight[plane] -= _eydz * norm;
}
}

if (dbias != 0) {
if (threadIdx.x == 0) {
dbias[plane] += _edz * norm;
}
}
}
}