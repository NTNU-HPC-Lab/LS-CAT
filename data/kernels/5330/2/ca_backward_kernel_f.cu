#include "hip/hip_runtime.h"
#include "includes.h"










/*
* Implementations
*/
__global__ void ca_backward_kernel_f(const float *dw, const float *t, const float *f, float *df, int num, int chn, int height, int width) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int sp = height * width;
int len = height + width - 1;
int plane = blockIdx.z;

if (x < width && y < height && plane < chn) {

for (int batch = 0; batch < num; ++batch) {

for (int i = 0; i < width; ++i) {
float _dw = dw[(batch * len + x) * sp + y*width + i];
float _t = t[(batch * chn + plane) * sp + y*width + i];
df[(batch * chn + plane) * sp + y*width + x] += _dw * _t;
}
for (int i = 0; i < height; ++i) {
if (i == y) continue;
int j = i>y ? y : y-1;

float _dw = dw[(batch * len + width + j) * sp + i*width + x];
float _t = t[(batch * chn + plane) * sp + i*width + x];
df[(batch * chn + plane) * sp + y*width + x] += _dw * _t;
}
}

}
}