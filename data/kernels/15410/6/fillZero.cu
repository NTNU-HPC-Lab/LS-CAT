#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillZero(int *c_red, int size)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = id; i < size; i+=stride)
{
c_red[i] = 0;
}
}