#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void countIndices(int *indices, unsigned int *histo, int size)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;

int min = blockIdx.x * blockDim.x;
int max = (blockIdx.x + 1) * blockDim.x;

extern __shared__ unsigned int tmp[];

tmp[threadIdx.x] = 0;

__syncthreads();

for (int i = threadIdx.x; i < size; i+=blockDim.x)
{
if (min <= indices[i] && indices[i] < max)
{
atomicAdd(&tmp[indices[i] - min], 1);
}
}

__syncthreads();

histo[id] = tmp[threadIdx.x];

return;
}