#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assignColIds(int* colIds, const int* colOffsets) {
int myId = blockIdx.x;
int start = colOffsets[myId];
int end = colOffsets[myId + 1];
for (int id = start + threadIdx.x; id < end; id += blockDim.x) {
colIds[id] = myId;
}
}