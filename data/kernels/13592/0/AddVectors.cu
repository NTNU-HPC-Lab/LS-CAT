#include "hip/hip_runtime.h"
#include "includes.h"
// Lab2_AddingTwoVectors.cu : Defines the entry point for the console application.
// Author: £ukasz Pawe³ Rabiec (259049)


#define SIZE 32

__global__ void AddVectors(int* a, int* b, int* c)
{
int tid = blockIdx.x;

if (tid < SIZE)
{
c[tid] = a[tid] + b[tid];
}

}