#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knn_assign_gmem_deinterleave1( uint32_t length, uint16_t k, uint32_t *neighbors) {
volatile uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
if (sample >= length) {
return;
}
if (sample % 2 == 1) {
for (int i = 0; i < k; i++) {
neighbors[sample * k + i] = neighbors[sample * 2 * k + i];
}
} else {
for (int i = 0; i < k; i++) {
neighbors[(length + sample) * k + k + i] = neighbors[sample * 2 * k + i];
}
}
}