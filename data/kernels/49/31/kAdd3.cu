#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd3(float* a, const float* b, const float* c, const unsigned int numEls, const float scaleA, const float scaleB, const float scaleC) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < numEls; i += numThreads) {
a[i] = scaleA * a[i] + scaleB * b[i] + scaleC * c[i];
}
}