#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const int srcWidth, const int srcHeight, const int tgtWidth, const int tgtHeight) {
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
const int numThreads = blockDim.x * gridDim.x;
//    const unsigned int numEls = tgtWidth * tgtHeight;
for (unsigned int i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
const int y = i / tgtWidth;
const int x = i % tgtWidth;
const int srcY = y % srcHeight;
const int srcX = x % srcWidth;
tgt[i] = src[srcY * srcWidth + srcX];
}
}