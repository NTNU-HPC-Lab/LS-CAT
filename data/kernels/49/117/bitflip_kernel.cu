#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitflip_kernel(float* M, int height, int row, int n) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
int off = blockDim.x * gridDim.x;
for (unsigned int i = idx; i < n; i += off){
M[i * height + row] = 1 - M[i * height + row];
}

}