#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogistic2(float* gData, float* target, unsigned int numElements) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
target[i] = 1 / (1 + expf(-gData[i]));
}