#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCopyInto(float* images, float* targets, const int imgSize, const int paddingSize, const int numImages) {
const int imgIdx = blockIdx.y * gridDim.x + blockIdx.x;
if (imgIdx < numImages) {
const int targetSize = imgSize + 2 * paddingSize;
images += imgIdx * imgSize * imgSize;
targets += imgIdx * targetSize * targetSize + MUL24(paddingSize, targetSize) + paddingSize;
for (int y = threadIdx.y; y < imgSize; y += 16) {
for (int x = threadIdx.x; x < imgSize; x += 16) {
targets[MUL24(y, targetSize) + x] = images[MUL24(y, imgSize) + x];
}
}
}
}