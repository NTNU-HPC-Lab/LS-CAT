#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDumbSumCols(float* mat, float* vec, unsigned int width, unsigned int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
mat += idx;
if (idx < width) {
float sum = 0;
for (int j = 0; j < height; j++) {
sum += *mat;
mat += width;
}
vec[idx] = sum;
}
}