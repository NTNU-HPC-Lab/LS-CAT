#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBiggerThanRowVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numThreads = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < width * height; i += numThreads) {
tgtMat[i] = mat[i] > vec[i % width];
}
}