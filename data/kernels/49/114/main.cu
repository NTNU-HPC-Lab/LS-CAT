#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "hessianKernelO.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_output = NULL;
hipMalloc(&d_output, XSIZE*YSIZE);
float *d_output_theta = NULL;
hipMalloc(&d_output_theta, XSIZE*YSIZE);
float *d_output_phi = NULL;
hipMalloc(&d_output_phi, XSIZE*YSIZE);
const float *d_gxx = NULL;
hipMalloc(&d_gxx, XSIZE*YSIZE);
const float *d_gxy = NULL;
hipMalloc(&d_gxy, XSIZE*YSIZE);
const float *d_gxz = NULL;
hipMalloc(&d_gxz, XSIZE*YSIZE);
const float *d_gyy = NULL;
hipMalloc(&d_gyy, XSIZE*YSIZE);
const float *d_gyz = NULL;
hipMalloc(&d_gyz, XSIZE*YSIZE);
const float *d_gzz = NULL;
hipMalloc(&d_gzz, XSIZE*YSIZE);
float sigma = 1;
int imageW = 1;
int imageH = 1;
int imageD = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
hessianKernelO<<<gridBlock,threadBlock>>>(d_output,d_output_theta,d_output_phi,d_gxx,d_gxy,d_gxz,d_gyy,d_gyz,d_gzz,sigma,imageW,imageH,imageD);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
hessianKernelO<<<gridBlock,threadBlock>>>(d_output,d_output_theta,d_output_phi,d_gxx,d_gxy,d_gxz,d_gyy,d_gyz,d_gzz,sigma,imageW,imageH,imageD);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
hessianKernelO<<<gridBlock,threadBlock>>>(d_output,d_output_theta,d_output_phi,d_gxx,d_gxy,d_gxz,d_gyy,d_gyz,d_gzz,sigma,imageW,imageH,imageD);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}