#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float computeDeterminant (float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22)
{
return e00*e11*e22-e00*e12*e21+e10*e21*e02-e10*e01*e22+e20*e01*e12-e20*e11*e02;
}
__global__ void hessianKernelO ( float *d_output, float *d_output_theta, float *d_output_phi, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD )
{
int n_blocks_per_width = imageW/blockDim.x;
int z = (int)ceilf(blockIdx.x/n_blocks_per_width);
int y = blockIdx.y*blockDim.y + threadIdx.y;
int x = (blockIdx.x - z*n_blocks_per_width)*blockDim.x + threadIdx.x;
int i = z*imageW*imageH + y*imageW + x;

// // //Brute force eigen-values computation
// http://en.wikipedia.org/wiki/Eigenvalue_algorithm
//Oliver K. Smith: Eigenvalues of a symmetric 3 × 3 matrix. Commun. ACM 4(4): 168 (1961)
float a0, b0, c0, d0, e0, f0;
a0 = -d_gxx[i]; b0 = -d_gxy[i]; c0 = -d_gxz[i];
d0 = -d_gyy[i]; e0 = -d_gyz[i]; f0 = -d_gzz[i];

float m = (a0+d0+f0)/3;
float q = computeDeterminant
(a0-m, b0, c0, b0, d0-m, e0, c0, e0, f0-m)/2;
float p = (a0-m)*(a0-m) + b0*b0 + c0*c0 + b0*b0 + (d0-m)*(d0-m) +
e0*e0 + c0*c0 + e0*e0 + (f0-m)*(f0-m);
p = p / 6;
float phi = 1.f/3.f*atan(sqrt(p*p*p-q*q)/q);
if(phi<0)
phi=phi+3.14159f/3;

float eig1 = m + 2*sqrt(p)*cos(phi);
float eig2 = m - sqrt(p)*(cos(phi) + sqrt(3.0f)*sin(phi));
float eig3 = m - sqrt(p)*(cos(phi) - sqrt(3.0f)*sin(phi));

if( (eig1 > eig2) & (eig1 > eig3))
d_output[i] = eig1*sigma*sigma;
if( (eig2 > eig1) & (eig2 > eig3))
d_output[i] = eig2*sigma*sigma;
if( (eig3 > eig2) & (eig3 > eig1))
d_output[i] = eig3*sigma*sigma;


// // Now it comes to compute the eigenvector
float l = d_output[i]/(sigma*sigma);
a0 = a0 - l;
d0 = d0 - l;
f0 = f0 - l;
float xv = b0*e0 - c0*d0;
float yv = e0*a0 - c0*b0;
float zv = d0*a0 - b0*b0;
float radius = sqrt(xv*xv+yv*yv+zv*zv);
float thetav = atan2(yv, xv);
float phiv = 0;
if(radius > 1e-6f)
phiv = acos( zv/radius);

d_output_theta[i] = thetav;
d_output_phi[i] = phiv;

}