#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float computeDeterminant (float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22)
{
return e00*e11*e22-e00*e12*e21+e10*e21*e02-e10*e01*e22+e20*e01*e12-e20*e11*e02;
}
__global__ void hessianKernel ( float *d_output, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD )
{
int n_blocks_per_width = imageW/blockDim.x;
int z = (int)ceilf(blockIdx.x/n_blocks_per_width);
int y = blockIdx.y*blockDim.y + threadIdx.y;
int x = (blockIdx.x - z*n_blocks_per_width)*blockDim.x + threadIdx.x;
int i = z*imageW*imageH + y*imageW + x;

// // //Brute force eigen-values computation
float a0, b0, c0, e0, f0, k0;
a0 = -d_gxx[i]; b0 = -d_gxy[i]; c0 = -d_gxz[i];
e0 = -d_gyy[i]; f0 = -d_gyz[i]; k0 = -d_gzz[i];


// http://en.wikipedia.org/wiki/Eigenvalue_algorithm
//Oliver K. Smith: Eigenvalues of a symmetric 3 × 3 matrix. Commun. ACM 4(4): 168 (1961)
float m = (a0+e0+k0)/3;
float q = computeDeterminant
(a0-m, b0, c0, b0, e0-m, f0, c0, f0, k0-m)/2;
float p = (a0-m)*(a0-m) + b0*b0 + c0*c0 + b0*b0 + (e0-m)*(e0-m) +
f0*f0 + c0*c0 + f0*f0 + (k0-m)*(k0-m);
p = p / 6;
float phi = 1.f/3.f*atan(sqrt(p*p*p-q*q)/q);
if(phi<0)
phi=phi+3.14159f/3;

float eig1 = m + 2*sqrt(p)*cos(phi);
float eig2 = m - sqrt(p)*(cos(phi) + sqrt(3.0f)*sin(phi));
float eig3 = m - sqrt(p)*(cos(phi) - sqrt(3.0f)*sin(phi));

if( (eig1 > eig2) & (eig1 > eig3))
d_output[i] = eig1*sigma*sigma;
if( (eig2 > eig1) & (eig2 > eig3))
d_output[i] = eig2*sigma*sigma;
if( (eig3 > eig2) & (eig3 > eig1))
d_output[i] = eig3*sigma*sigma;
}