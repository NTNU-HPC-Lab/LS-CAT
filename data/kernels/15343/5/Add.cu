#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float *a, float *b, float *c)
{
int Id = threadIdx.x + blockDim.x * blockIdx.x;
if (Id < N) {
a[Id] = threadIdx.x;
b[Id] = blockIdx.x;
c[Id] = Id;
}
}