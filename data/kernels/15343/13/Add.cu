#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float *a, float *b, float *c)
{
int Id = threadIdx.x + blockDim.x * blockIdx.x;
printf("(%d, %d, %d) ", threadIdx.x, blockDim.x, blockIdx.x);
printf("hilo: %d, ", Id);
//solo trabajan los N hilos
if (Id < N) {
c[Id] = a[Id] * b[Id];
}
}