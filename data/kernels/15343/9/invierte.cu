#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void invierte(float *a, float *b) {
int id = threadIdx.x;
//int id = threadIdx.x + blockDim.x * blockIdx.x;// para n-bloques de 1 hilo

if (id < N)
{
b[id] = a[N-id];
}
}