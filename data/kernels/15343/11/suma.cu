#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void suma(float *A, float *B, float *C)
{
//indice de las columnas
int columna = threadIdx.x;
//indice de las filas
int fila = threadIdx.y;
//indice lineal
int Id = columna + fila * blockDim.x;
//sumamos cada elemento
C[Id] = A[Id] + B[Id];
}