#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solve_GPU(int a, int b, int c ,int *x1, int *x2)
{
int raiz = powf(b, 2) - (4 * a * c);
int i = -b / 2 * a;
int j = 2 * a;

*x1 = i + sqrtf(raiz) / j;
*x2 = i - sqrtf(raiz) / j;
}