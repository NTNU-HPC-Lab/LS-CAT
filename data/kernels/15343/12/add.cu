#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *A, float *C)
{
int columna = threadIdx.x;
//indice de las filas
int fila = threadIdx.y;
//indice lineal
int Id = columna + fila * blockDim.x;

int id1 = (columna - 1) + fila * blockDim.x;
int id2 = (columna + 1) + fila * blockDim.x;
int id3 = columna + (fila - 1) * blockDim.x;
int id4 = columna + (fila + 1) * blockDim.x;

if ((fila > 0 && fila < N - 1) && (columna > 0 && columna < N - 1)) {

C[Id] = A[id1] + A[id2] + A[id3] + A[id4];
}
else
{
C[Id] = A[Id];
}
}