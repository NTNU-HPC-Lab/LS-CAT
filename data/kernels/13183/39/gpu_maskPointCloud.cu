#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_maskPointCloud(float4* verts, const int width, const int height, const int* mask) {

const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x >= width || y >= height)
return;

const int index = x + y*width;

int m = mask[index];
if (m == 0) {
verts[index].w = -1;
}

}