#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_imageSqrt(float * out, const float * in, const int width, const int height) {

const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x >= width || y >= height) {
return;
}

int index = x + y*width;
out[index] = sqrtf(in[index]);

}