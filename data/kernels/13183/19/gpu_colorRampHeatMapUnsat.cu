#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_colorRampHeatMapUnsat(uchar3 * colored, const float * vals, const int width, const int height, const float minVal, const float maxVal) {

const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x >= width || y >= height) {
return;
}

const int index = x + y*width;
uchar3 & imgVal = colored[index];

if (isnan(vals[index])) {
imgVal = make_uchar3(255,255,255);
return;
}

const float normVal = fmaxf(0,fminf((vals[index] - minVal)/(maxVal-minVal),1));

const float t = normVal == 1.0 ? 1.0 : fmodf(normVal,0.25)*4;
uchar3 a, b;
if (normVal < 0.25) { b = make_uchar3(32,191,139); a = make_uchar3(0x18,0x62,0x93); }
else if (normVal < 0.5) { b = make_uchar3(241,232,137); a = make_uchar3(32,191,139); }
else if (normVal < 0.75) { b = make_uchar3(198,132,63); a = make_uchar3(241,232,137); }
else { b = make_uchar3(0xc0,0x43,0x36); a = make_uchar3(198,132,63); }
imgVal = make_uchar3((1-t)*a.x + t*b.x,
(1-t)*a.y + t*b.y,
(1-t)*a.z + t*b.z);

}