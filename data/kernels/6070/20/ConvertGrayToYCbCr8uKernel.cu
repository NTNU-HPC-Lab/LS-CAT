#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvertGrayToYCbCr8uKernel(const uint8_t *input, uint8_t *output, unsigned int total_pixels) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= total_pixels) {
return;
}

const uint8_t pixel_in = input[idx];
const unsigned int C = 3;
uint8_t* pixel_out = &output[idx * C];
pixel_out[0] = pixel_in;
pixel_out[1] = 128;
pixel_out[2] = 128;
}