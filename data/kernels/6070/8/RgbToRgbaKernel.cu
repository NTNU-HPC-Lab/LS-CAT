#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RgbToRgbaKernel(const uint8_t *__restrict__ input, uint8_t *__restrict__ output, size_t pitch, size_t width_px, size_t height) {
constexpr size_t in_channels = 3, out_channels = 4;
size_t x = threadIdx.x + blockIdx.x * blockDim.x;
size_t y = threadIdx.y + blockIdx.y * blockDim.y;
if (x >= width_px || y >= height) return;
size_t in_idx = in_channels * x + in_channels * width_px * y;
size_t out_idx = out_channels * x + pitch * y;
output[out_idx] = input[in_idx];
output[out_idx + 1] = input[in_idx + 1];
output[out_idx + 2] = input[in_idx + 2];
output[out_idx + 3] = 255;
}