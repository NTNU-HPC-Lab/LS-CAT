#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_broadcast_16(int * in, int *out)
{
int x = in[threadIdx.x];
int y = __shfl_sync(0xffffffff, x, 3, 16);
out[threadIdx.x] = y;
}