#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_arrays_2Dgrid_2Dblock(float* a, float* b, float *c, int nx, int ny)
{
int gidx = blockIdx.x * blockDim.x + threadIdx.x;
int gidy = blockIdx.y*blockDim.y + threadIdx.y;

int gid = gidy * nx + gidx;

if(gidx < nx && gidy < ny)
c[gid] = a[gid] + b[gid];
}