#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_read_row_write_column(int * mat, int * transpose, int nx, int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
transpose[ix * ny + iy] = mat[iy * nx + ix];
}
}