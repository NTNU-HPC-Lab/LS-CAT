#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_interleaved_unrolling_blocks2_1(int * input, int * temp, int size)
{
int tid = threadIdx.x;

//start index for this thread
int index = blockDim.x * blockIdx.x * 2 + threadIdx.x;

//local index for this block
int * i_data = input + blockDim.x * blockIdx.x * 2;

//unrolling two blocks
if ((index + blockDim.x)< size)
{
input[index] += input[index + blockDim.x];
}

__syncthreads();

for (int offset = blockDim.x / 2; offset > 0;
offset = offset / 2)
{
if (tid < offset)
{
i_data[tid] += i_data[tid + offset];
}

__syncthreads();
}

if (tid == 0)
{
temp[blockIdx.x] = i_data[0];
}
}