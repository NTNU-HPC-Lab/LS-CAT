#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_neighbored_pairs_improved_1( int * int_array, int * temp_array, int size)
{
int tid = threadIdx.x;
int gid = blockDim.x * blockIdx.x + threadIdx.x;

//local data block pointer
int * i_data = int_array + blockDim.x * blockIdx.x;

if (gid > size)
return;

for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
{
int index = 2 * offset * tid;

if (index < blockDim.x)
{
i_data[index] += i_data[index + offset];
}

__syncthreads();
}

if (tid == 0)
{
temp_array[blockIdx.x] = int_array[gid];
}
}