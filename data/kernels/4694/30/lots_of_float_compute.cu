#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lots_of_float_compute(float *inputs, int N, size_t niters, float *outputs)
{
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
size_t nthreads = gridDim.x * blockDim.x;

for (; tid < N; tid += nthreads)
{
size_t iter;
float val = inputs[tid];

for (iter = 0; iter < niters; iter++)
{
val = (val + 5.0f) - 101.0f;
val = (val / 3.0f) + 102.0f;
val = (val + 1.07f) - 103.0f;
val = (val / 1.037f) + 104.0f;
val = (val + 3.00f) - 105.0f;
val = (val / 0.22f) + 106.0f;
}

outputs[tid] = val;
}
}