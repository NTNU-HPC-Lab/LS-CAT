#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compact_1D_array( int * input, int * output, int * output_index_array, int array_size)
{
int gid = blockDim.x*blockIdx.x + threadIdx.x;

//TO DO handle when gid ==0
//this is very unefficient in memory management
if (gid > 0 && gid < array_size)
{
if (output_index_array[gid] != output_index_array[gid - 1])
{
//printf("gid : %d , index :%d , value : %d, prev_value : %d \n",gid, output_index_array[gid], input[gid], input[gid-1]);
output[output_index_array[gid]] = input[gid-1];
}
}
}