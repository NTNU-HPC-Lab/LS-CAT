#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_unroll4_row(int * mat, int * transpose, int nx, int ny)
{
int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int ti = iy * nx + ix;
int to = ix * ny + iy;

if (ix + 3 * blockDim.x < nx && iy < ny)
{
transpose[to]						= mat[ti];
transpose[to + ny*blockDim.x]		= mat[ti + blockDim.x];
transpose[to + ny * 2 * blockDim.x] = mat[ti + 2 * blockDim.x];
transpose[to + ny * 3 * blockDim.x] = mat[ti + 3 * blockDim.x];
}
}