#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_smem(int * in, int* out, int nx, int ny)
{
__shared__ int tile[BDIMY][BDIMX];

//input index
int ix, iy, in_index;

//output index
int i_row, i_col, _1d_index, out_ix, out_iy, out_index;

//ix and iy calculation for input index
ix = blockDim.x * blockIdx.x + threadIdx.x;
iy = blockDim.y * blockIdx.y + threadIdx.y;

//input index
in_index = iy * nx + ix;

//1D index calculation fro shared memory
_1d_index = threadIdx.y * blockDim.x + threadIdx.x;

//col major row and col index calcuation
i_row = _1d_index / blockDim.y;
i_col = _1d_index % blockDim.y;

//coordinate for transpose matrix
out_ix = blockIdx.y * blockDim.y + i_col;
out_iy = blockIdx.x * blockDim.x + i_row;

//output array access in row major format
out_index = out_iy * ny + out_ix;

if (ix < nx && iy < ny)
{
//load from in array in row major and store to shared memory in row major
tile[threadIdx.y][threadIdx.x] = in[in_index];

//wait untill all the threads load the values
__syncthreads();

out[out_index] = tile[i_col][i_row];
}
}