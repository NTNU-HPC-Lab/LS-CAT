#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void read_stride_write_coaleased_mat_trans(float* input, float* output, const int nx, const int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
output[iy*nx + ix] = input[ix*ny + iy];
}
}