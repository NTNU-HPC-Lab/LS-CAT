#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_read_column_write_row(int * mat, int * transpose, int nx, int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
transpose[iy * nx + ix] = mat[ix * ny + iy];
}
}