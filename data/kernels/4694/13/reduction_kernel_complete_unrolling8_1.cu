#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel_complete_unrolling8_1(int * input, int * temp, int size)
{
int tid = threadIdx.x;
int index = blockDim.x * blockIdx.x * 8 + threadIdx.x;

int * i_data = input + blockDim.x * blockIdx.x * 8;

if ((index + 7 * blockDim.x) < size)
{
int a1 = input[index];
int a2 = input[index + blockDim.x];
int a3 = input[index + 2 * blockDim.x];
int a4 = input[index + 3 * blockDim.x];
int a5 = input[index + 4 * blockDim.x];
int a6 = input[index + 5 * blockDim.x];
int a7 = input[index + 6 * blockDim.x];
int a8 = input[index + 7 * blockDim.x];

input[index] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
}

__syncthreads();

//complete unrolling manually

//if the block dim == 1024
if (blockDim.x == 1024 && tid < 512)
i_data[tid] += i_data[tid + 512];
__syncthreads();

if (blockDim.x >= 512 && tid < 256)
i_data[tid] += i_data[tid + 256];
__syncthreads();

if (blockDim.x >= 256 && tid < 128)
i_data[tid] += i_data[tid + 128];
__syncthreads();

if (blockDim.x >= 128 && tid < 64)
i_data[tid] += i_data[tid + 64];
__syncthreads();


// warp unrolling
if (tid < 32)
{
volatile int * vsmem = i_data;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid + 8];
vsmem[tid] += vsmem[tid + 4];
vsmem[tid] += vsmem[tid + 2];
vsmem[tid] += vsmem[tid + 1];
}

if (tid == 0)
{
temp[blockIdx.x] = i_data[0];
}
}