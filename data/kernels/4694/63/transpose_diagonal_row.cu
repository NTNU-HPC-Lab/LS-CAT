#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_diagonal_row(int * mat, int * transpose, int nx, int ny)
{
int blk_x = blockIdx.x;
int blk_y = (blockIdx.x + blockIdx.y) % gridDim.x;

int ix = blockIdx.x * blk_x + threadIdx.x;
int iy = blockIdx.y * blk_y + threadIdx.y;

if (ix < nx && iy < ny)
{
transpose[ix * ny + iy] = mat[iy * nx + ix];
}
}