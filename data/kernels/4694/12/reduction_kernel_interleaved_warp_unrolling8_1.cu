#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel_interleaved_warp_unrolling8_1(int * input, int * temp_array, int size)
{
int tid = threadIdx.x;

//element index for this thread
int index = blockDim.x * blockIdx.x * 8 + threadIdx.x;

//local data pointer
int * i_data = input + blockDim.x * blockIdx.x * 8;

if ((index + 7 * blockDim.x) < size)
{
int a1 = input[index];
int a2 = input[index + blockDim.x];
int a3 = input[index + 2 * blockDim.x];
int a4 = input[index + 3 * blockDim.x];
int a5 = input[index + 4 * blockDim.x];
int a6 = input[index + 5 * blockDim.x];
int a7 = input[index + 6 * blockDim.x];
int a8 = input[index + 7 * blockDim.x];

input[index] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
}

__syncthreads();

for (int offset = blockDim.x / 2; offset >= 64;
offset = offset / 2)
{
if (tid < offset)
{
i_data[tid] += i_data[tid + offset];
}
__syncthreads();
}

if (tid < 32)
{
volatile int * vsmem = i_data;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid + 8];
vsmem[tid] += vsmem[tid + 4];
vsmem[tid] += vsmem[tid + 2];
vsmem[tid] += vsmem[tid + 1];
}

if (tid == 0)
{
temp_array[blockIdx.x] = i_data[0];
}
}