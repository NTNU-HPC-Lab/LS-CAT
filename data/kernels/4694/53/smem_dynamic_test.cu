#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smem_dynamic_test(int * in, int * out, int size)
{
int tid = threadIdx.x;
int gid = blockIdx.x * blockDim.x + threadIdx.x;

extern __shared__ int smem[];

if (gid < size)
{
smem[tid] = in[gid];
out[gid] = smem[tid];
}
}