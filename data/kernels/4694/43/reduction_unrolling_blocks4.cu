#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_unrolling_blocks4(int * input, int * temp, int size)
{
int tid = threadIdx.x;

int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;

int index = BLOCK_OFFSET + tid;

int * i_data = input + BLOCK_OFFSET;

if ((index + 3 * blockDim.x) < size)
{
int a1 = input[index];
int a2 = input[index + blockDim.x];
int a3 = input[index+ 2* blockDim.x];
int a4 = input[index+ 3 *blockDim.x];
input[index] = a1 + a2 + a3 + a4;
}

__syncthreads();

for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2)
{
if (tid < offset)
{
i_data[tid] += i_data[tid + offset];
}

__syncthreads();
}

if (tid == 0)
{
temp[blockIdx.x] = i_data[0];
}
}