#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_transpose_k1(float* input,float* output,const int nx, const int ny)
{
int gid = blockDim.x * blockIdx.x + threadIdx.x;
int offset = threadIdx.x*blockDim.x;
//printf("gid : %d , offset : %d , index : %d ,value : %f \n", gid, offset, offset + blockIdx.x,input[offset + blockIdx.x]);
output[gid] = input[offset + blockIdx.x];
}