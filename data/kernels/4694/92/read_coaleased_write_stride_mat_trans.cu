#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void read_coaleased_write_stride_mat_trans(float* input, float* output, const int nx, const int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
output[ix*ny + iy] = input[iy*nx + ix];
}
}