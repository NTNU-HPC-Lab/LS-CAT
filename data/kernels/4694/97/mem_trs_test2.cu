#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mem_trs_test2(int * input, int size)
{
int gid = blockIdx.x * blockDim.x + threadIdx.x;

if(gid < size)
printf("tid : %d , gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
}