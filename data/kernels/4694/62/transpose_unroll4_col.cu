#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_unroll4_col(int * mat, int * transpose, int nx, int ny)
{
int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

int ti = iy * nx + ix;
int to = ix * ny + iy;

if (ix + 3 * blockDim.x < nx && iy < ny)
{
transpose[ti] = mat[to];
transpose[ti + blockDim.x] = mat[to + blockDim.x*ny];
transpose[ti + 2 * blockDim.x] = mat[to + 2 * blockDim.x*ny];
transpose[ti + 3 * blockDim.x] = mat[to + 3 * blockDim.x*ny];
}
}