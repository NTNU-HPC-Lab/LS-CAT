#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  register_usage_test(int * results, int size)
{
int gid = blockDim.x * blockIdx.x + threadIdx.x;

int x1 = 3465;
int x2 = 1768;
int x3 = 453;
int x7 = 3465;
int x5 = 1768;
int x6 = 453;
int x4 = x1 + x2 + x3 + x7 + x5 + x6;

if (gid < size)
{
results[gid] =  x4;
}
}