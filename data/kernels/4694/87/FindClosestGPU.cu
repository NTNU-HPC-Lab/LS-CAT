#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindClosestGPU(float3* points, int* indices, int count)
{
if (count <= 1) return;
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < count)
{
float3 thisPoint = points[idx]; // every thread takes its own point
float smallestDistSoFar = 3.40282e38f; // almost the biggest possible floating point value
int smallestIdxSoFar = -1;

// run through the list of all other points
for (int i = 0; i < count; i++)
{
if (i == idx) continue;
float dist = (thisPoint.x - points[i].x)*(thisPoint.x - points[i].x);
dist += (thisPoint.y - points[i].y)*(thisPoint.y - points[i].y);
dist += (thisPoint.z - points[i].z)*(thisPoint.z - points[i].z);
if (dist < smallestDistSoFar)
{
smallestDistSoFar = dist;
smallestIdxSoFar = i;
}
}
indices[idx] = smallestIdxSoFar;
}
}