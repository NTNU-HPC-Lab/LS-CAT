#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void misaligned_read_test(float* a, float* b, float *c, int size, int offset)
{
int gid = blockIdx.x * blockDim.x + threadIdx.x;
int k = gid + offset;

if (k < size)
c[gid] = a[k]+ b[k];

//c[gid] = a[gid];
}