#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_rows(float* input, float* output, const int nx, const int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
output[iy*ny + ix] = input[iy*nx + ix];
}
}