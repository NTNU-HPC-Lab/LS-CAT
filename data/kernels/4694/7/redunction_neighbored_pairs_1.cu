#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void redunction_neighbored_pairs_1(int * input, int * temp, int size)
{
int tid = threadIdx.x;
int gid = blockDim.x * blockIdx.x + threadIdx.x;

if (gid > size)
return;

for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
{
if (tid % (2 * offset) == 0)
{
input[gid] += input[gid + offset];
}

__syncthreads();
}

if (tid == 0)
{
temp[blockIdx.x] = input[gid];
}
}