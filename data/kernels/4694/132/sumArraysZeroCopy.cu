#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopy(int *A, int *B, int *C, const int N)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i] = A[i] + B[i];
}