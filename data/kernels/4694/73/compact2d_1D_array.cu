#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compact2d_1D_array(int * input, int * output, int * output_column_index_array, int * output_row_index_array, int * prev_output_index_array,int* auxiliry_array, int array_size)
{
int gid = blockDim.x*blockIdx.x + threadIdx.x;

//TO DO handle when gid ==0
//this is very unefficient in memory management
if (gid > 0 && gid < array_size)
{
printf("gid : %d , index :%d , value : %d, prev_value : %d \n",gid, prev_output_index_array[gid], input[gid], input[gid-1]);
if (prev_output_index_array[gid] != prev_output_index_array[gid - 1])
{
//printf("gid : %d , index :%d , value : %d, prev_value : %d \n",gid, output_index_array[gid], input[gid], input[gid-1]);
output[prev_output_index_array[gid]] = input[gid - 1];
output_column_index_array[prev_output_index_array[gid]] = (gid - 1)% blockDim.x;
}

int colum_index = gid / (blockDim.x  - 1);
int condition = gid % (blockDim.x - 1);

if (condition == 0)
{
printf("column index : %d --- row length : %d \n", condition, prev_output_index_array[gid]);
if (gid == 0)
{
output_row_index_array[0] = 0;
}
else
{
output_row_index_array[colum_index] = prev_output_index_array[gid];
//output_row_index_array[colum_index] = auxiliry_array[colum_index];
}
}
}
}