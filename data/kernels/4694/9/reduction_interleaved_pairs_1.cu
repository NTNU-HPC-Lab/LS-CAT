#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_interleaved_pairs_1(int * int_array, int * temp_array, int size)
{
int tid = threadIdx.x;
int gid = blockDim.x * blockIdx.x + threadIdx.x;

if (gid > size)
return;

for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2)
{
if (tid < offset)
{
int_array[gid] += int_array[gid + offset];
}

__syncthreads();
}

if (tid == 0)
{
temp_array[blockIdx.x] = int_array[gid];
}
}