#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void misaligned_read_unrolled4(int* a, int* b, int *c, int size, int offset)
{
int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
int k = i + offset;

if (k + 3 * blockDim.x < size)
{
c[i] = a[k] + b[k];
c[i + blockDim.x] = a[k + blockDim.x] + b[k + blockDim.x];
c[i + 2* blockDim.x] = a[k + 2 * blockDim.x] + b[k + 2 *blockDim.x];
c[i + 3* blockDim.x] = a[k + 3* blockDim.x] + b[k + 3* blockDim.x];
}
}