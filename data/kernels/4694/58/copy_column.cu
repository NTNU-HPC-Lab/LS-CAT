#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_column(int * mat, int * transpose, int nx, int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
transpose[ix * ny + iy] = mat[ix * ny + iy];
}
}