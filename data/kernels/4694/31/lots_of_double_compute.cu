#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
size_t nthreads = gridDim.x * blockDim.x;

for (; tid < N; tid += nthreads)
{
size_t iter;
double val = inputs[tid];

for (iter = 0; iter < niters; iter++)
{
val = (val + 5.0) - 101.0;
val = (val / 3.0) + 102.0;
val = (val + 1.07) - 103.0;
val = (val / 1.037) + 104.0;
val = (val + 3.00) - 105.0;
val = (val / 0.22) + 106.0;
}

outputs[tid] = val;
}
}