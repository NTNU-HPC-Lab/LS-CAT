#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_xor_butterfly(int * in, int *out)
{
int x = in[threadIdx.x];
int y = __shfl_xor_sync(0xffffffff, x, 1, 32);
out[threadIdx.x] = y;
}