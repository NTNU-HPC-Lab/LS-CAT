#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_columns(float* input, float* output, const int nx, const int ny)
{
int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;

if (ix < nx && iy < ny)
{
output[ix*ny + iy] = input[ix*ny + iy];
}
}