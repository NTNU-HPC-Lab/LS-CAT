#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_smem_pad_unrolling(int * in, int* out, int nx, int ny)
{
__shared__ int tile[BDIMY * (2 * BDIMX + IPAD)];

//input index
int ix, iy, in_index;

//output index
int i_row, i_col, _1d_index, out_ix, out_iy, out_index;

//ix and iy calculation for input index
ix = 2 * blockDim.x * blockIdx.x + threadIdx.x;
iy = blockDim.y * blockIdx.y + threadIdx.y;

//input index
in_index = iy * nx + ix;

//1D index calculation fro shared memory
_1d_index = threadIdx.y * blockDim.x + threadIdx.x;

//col major row and col index calcuation
i_row = _1d_index / blockDim.y;
i_col = _1d_index % blockDim.y;

//coordinate for transpose matrix
out_ix = blockIdx.y * blockDim.y + i_col;
out_iy = 2 * blockIdx.x * blockDim.x + i_row;

//output array access in row major format
out_index = out_iy * ny + out_ix;

if (ix < nx && iy < ny)
{
int row_idx = threadIdx.y * (2 * blockDim.x + IPAD) + threadIdx.x;

//load from in array in row major and store to shared memory in row major
tile[row_idx] = in[in_index];
tile[row_idx+ BDIMX] = in[in_index + BDIMX];

//wait untill all the threads load the values
__syncthreads();

int col_idx = i_col * (2 * blockDim.x + IPAD) + i_row;

out[out_index] = tile[col_idx];
out[out_index + ny* BDIMX] = tile[col_idx + BDIMX];
}
}