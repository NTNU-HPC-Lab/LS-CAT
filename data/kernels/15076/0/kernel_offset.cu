#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_offset(int *key, int *idx, int *offset, int size) {

int idxX = threadIdx.x + blockIdx.x*blockDim.x;

if(idxX == 0) {
offset[1] = 0;
}
else if(idxX < size) {
int keyVal = key[idxX];
int keyValPrev = key[idxX-1];
if(keyVal != keyValPrev) {
offset[keyVal+1] = idxX;
}
}
if(idxX == size-1) {
int keyVal = key[idxX];
offset[0] = keyVal+1;
offset[keyVal+2] = size;
}
}