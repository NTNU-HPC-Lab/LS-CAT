#include "hip/hip_runtime.h"
#include "includes.h"



#define TIME                5.
#define TIME_STEP           .1

#define STEP                1.
#define K                   TIME_STEP / SQUARE(STEP)

#define SQUARE(x)           (x * x)
#define HANDLE_ERROR(err)   (HandleError(err, __FILE__, __LINE__))


__global__ void Kernel(double * device, const uint size)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;

if (i == 0) {
device[i] = .0;
} else if (i == size - 1) {
device[size - 1] = device[size - 2] + 5 * STEP;
} else if (i < size) {
device[i] = (device[i + 1] - 2 * device[i] + device[i - 1]) * K + device[i];
}
}