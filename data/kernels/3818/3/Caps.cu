#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Caps(char *c, int *b)
{
int tid = blockIdx.x;
if (tid < N)
{
if (b[tid] == 1)
{
int ascii = (int)c[tid];
ascii -= 32;
c[tid] = (char)ascii;
}
}

}