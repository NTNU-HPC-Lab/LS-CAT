#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
//blockIdx is the value of the block index for whichever block is running the code
int tid = blockIdx.x;//handle the data at this index
//blockIdx has 2 dimensions; x and y. We only need one dimension
if(tid < N)
c[tid] = a[tid] + b[tid];
}