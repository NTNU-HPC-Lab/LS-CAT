#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Turtle(int *v1, int *v2, int *lead)
{
if (*v1 >= *v2)
{
printf("%d, %d, %d", -1, -1,-1);
}
else
{
printf("%d, %d, %d\n", *v1, *v2, *lead);
double _result = ((float)*lead)/(((float)*v2)-((float)*v1));
int h = _result;
int m = _result * 60 - h*60;
int s = (_result * 3600) -m*60;
printf("%.3f\n",_result);
printf("%d, %d, %d\n", h, m, s);

}
}