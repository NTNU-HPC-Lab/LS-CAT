#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_rndrd_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
// we don't want completely random writes here, since the performance would be awful
// instead, let each warp move around randomly, but keep the warp coalesced on 128B-aligned
//  accesses
int errors = 0;
for(size_t j = 0; j < reps; j++) {
// starting point is naturally aligned
size_t p = blockIdx.x * blockDim.x + threadIdx.x;
// if we start outside the block, sit this out (just to keep small runs from crashing)
if(p >= elements) break;

// quadratic stepping via "acceleration" and "velocity"
size_t a = 548191;
size_t v = 24819 + (p >> 5);  // velocity has to be different for each warp

for(size_t i = 0; i < steps; i += 4) {
// delta is multiplied by 32 elements so warp stays converged (velocity is the
//  same for all threads in the warp)
// manually unroll loop to get multiple loads in flight per thread
size_t p0 = p;
p = (p + (v << 5)) % elements;
v = (v + a) % elements;
size_t p1 = p;
p = (p + (v << 5)) % elements;
v = (v + a) % elements;
size_t p2 = p;
p = (p + (v << 5)) % elements;
v = (v + a) % elements;
size_t p3 = p;
p = (p + (v << 5)) % elements;
v = (v + a) % elements;

int v0 = buffer[p0];
int v1 = buffer[p1];
int v2 = buffer[p2];
int v3 = buffer[p3];

if(v0 != p1) errors++;
if(v1 != p2) errors++;
if(v2 != p3) errors++;
if(v3 != p) errors++;
}
}
if((errors > 0) && (reps > elements))
buffer[0] = errors;
}