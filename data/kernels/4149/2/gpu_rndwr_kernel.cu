#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_rndwr_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
// we don't want completely random writes here, since the performance would be awful
// instead, let each warp move around randomly, but keep the warp coalesced on 128B-aligned
//  accesses
for(size_t j = 0; j < reps; j++) {
// starting point is naturally aligned
size_t p = blockIdx.x * blockDim.x + threadIdx.x;
// if we start outside the block, sit this out (just to keep small runs from crashing)
if(p >= elements) break;

// quadratic stepping via "acceleration" and "velocity"
size_t a = 548191;
size_t v = 24819 + (p >> 5);  // velocity has to be different for each warp

for(size_t i = 0; i < steps; i++) {
size_t prev = p;
// delta is multiplied by 32 elements so warp stays converged (velocity is the
//  same for all threads in the warp)
p = (p + (v << 5)) % elements;
v = (v + a) % elements;
buffer[prev] = p;
}
}
}