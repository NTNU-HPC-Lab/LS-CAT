#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_latency_setup_kernel(int *buffer, size_t delta, size_t elements)
{
size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
size_t step = blockDim.x * gridDim.x;
while(ofs < elements) {
size_t tgt = ofs + delta;
while(tgt >= elements)
tgt -= elements;
buffer[ofs] = tgt;
ofs += step;
}
}