#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_seqrd_kernel(int *buffer, size_t reps, size_t elements)
{
int errors = 0;
for(size_t j = 0; j < reps; j++) {
size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
size_t step = blockDim.x * gridDim.x;
while(ofs < elements) {
// manually unroll loop to get multiple loads in flight per thread
int val1 = buffer[ofs];
ofs += step;
int val2 = (ofs < elements) ? buffer[ofs] : 0;
ofs += step;
int val3 = (ofs < elements) ? buffer[ofs] : 0;
ofs += step;
int val4 = (ofs < elements) ? buffer[ofs] : 0;
ofs += step;
// now check result of all the reads
if(val1 != 0) errors++;
if(val2 != 0) errors++;
if(val3 != 0) errors++;
if(val4 != 0) errors++;
}
}
if(errors > 0)
buffer[0] = errors;
}