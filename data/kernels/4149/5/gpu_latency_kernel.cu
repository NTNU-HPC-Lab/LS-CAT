#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_latency_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
int errors = 0;
// this is done with just a single thread
for(size_t j = 0; j < reps; j++) {
int p = j & 31;

for(size_t i = 0; i < steps; i++) {
int next = buffer[p];

if((next >= 0) && (next < elements)) {
p = next;
} else {
printf("%d -> %d\n", p, next);
p = 0;
errors++;
}
}
}
if((errors > 0) && (reps > elements))
buffer[0] = errors;
}