#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_seqwr_kernel(int *buffer, size_t reps, size_t elements)
{
for(size_t j = 0; j < reps; j++) {
size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
size_t step = blockDim.x * gridDim.x;
while(ofs < elements) {
buffer[ofs] = 0;
ofs += step;
}
}
}