#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "make_pillar_index_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *dev_pillar_count_histo = NULL;
hipMalloc(&dev_pillar_count_histo, XSIZE*YSIZE);
int *dev_counter = NULL;
hipMalloc(&dev_counter, XSIZE*YSIZE);
int *dev_pillar_count = NULL;
hipMalloc(&dev_pillar_count, XSIZE*YSIZE);
int *dev_x_coors = NULL;
hipMalloc(&dev_x_coors, XSIZE*YSIZE);
int *dev_y_coors = NULL;
hipMalloc(&dev_y_coors, XSIZE*YSIZE);
float *dev_x_coors_for_sub = NULL;
hipMalloc(&dev_x_coors_for_sub, XSIZE*YSIZE);
float *dev_y_coors_for_sub = NULL;
hipMalloc(&dev_y_coors_for_sub, XSIZE*YSIZE);
float *dev_num_points_per_pillar = NULL;
hipMalloc(&dev_num_points_per_pillar, XSIZE*YSIZE);
int *dev_sparse_pillar_map = NULL;
hipMalloc(&dev_sparse_pillar_map, XSIZE*YSIZE);
const int max_pillars = 1;
const int max_points_per_pillar = 1;
const int GRID_X_SIZE = 1;
const float PILLAR_X_SIZE = 1;
const float PILLAR_Y_SIZE = 1;
const int NUM_INDS_FOR_SCAN = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
make_pillar_index_kernel<<<gridBlock,threadBlock>>>(dev_pillar_count_histo,dev_counter,dev_pillar_count,dev_x_coors,dev_y_coors,dev_x_coors_for_sub,dev_y_coors_for_sub,dev_num_points_per_pillar,dev_sparse_pillar_map,max_pillars,max_points_per_pillar,GRID_X_SIZE,PILLAR_X_SIZE,PILLAR_Y_SIZE,NUM_INDS_FOR_SCAN);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
make_pillar_index_kernel<<<gridBlock,threadBlock>>>(dev_pillar_count_histo,dev_counter,dev_pillar_count,dev_x_coors,dev_y_coors,dev_x_coors_for_sub,dev_y_coors_for_sub,dev_num_points_per_pillar,dev_sparse_pillar_map,max_pillars,max_points_per_pillar,GRID_X_SIZE,PILLAR_X_SIZE,PILLAR_Y_SIZE,NUM_INDS_FOR_SCAN);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
make_pillar_index_kernel<<<gridBlock,threadBlock>>>(dev_pillar_count_histo,dev_counter,dev_pillar_count,dev_x_coors,dev_y_coors,dev_x_coors_for_sub,dev_y_coors_for_sub,dev_num_points_per_pillar,dev_sparse_pillar_map,max_pillars,max_points_per_pillar,GRID_X_SIZE,PILLAR_X_SIZE,PILLAR_Y_SIZE,NUM_INDS_FOR_SCAN);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}