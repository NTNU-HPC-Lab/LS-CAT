#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int row = blockIdx.y;
int col = blockIdx.z;

if (row < 3 && col < 6) {
double *icov0 = inverse_covariance + row * 3 * voxel_num;
double *icov1 = icov0 + voxel_num;
double *icov2 = icov1 + voxel_num;
double *cov_dxd_pi_tmp = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
double *pg_tmp0 = point_gradients + col * valid_points_num;
double *pg_tmp1 = pg_tmp0 + 6 * valid_points_num;
double *pg_tmp2 = pg_tmp1 + 6 * valid_points_num;

for (int i = id; i < valid_points_num; i += stride) {
double pg0 = pg_tmp0[i];
double pg1 = pg_tmp1[i];
double pg2 = pg_tmp2[i];

for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
int vid = voxel_id[j];

cov_dxd_pi_tmp[j] = icov0[vid] * pg0 + icov1[vid] * pg1 + icov2[vid] * pg2;
}
}
}
}