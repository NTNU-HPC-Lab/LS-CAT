#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (index >= n*size*batch) return;
int i = index % size;
index /= size;
int j = index % n;
index /= n;
int k = index;

output[(k*n+j)*size + i] += biases[j];
}