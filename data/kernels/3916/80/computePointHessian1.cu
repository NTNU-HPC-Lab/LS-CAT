#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointHessian1(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph124, double *ph134, double *ph144, double *ph154, double *ph125, double *ph164, double *ph135, double *ph174, double *ph145)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
__shared__ double h_ang[18];

if (threadIdx.x < 18) {
h_ang[threadIdx.x] = dh_ang[18 + threadIdx.x];
}

__syncthreads();

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];

//Orignal coordinates
double o_x = static_cast<double>(x[pid]);
double o_y = static_cast<double>(y[pid]);
double o_z = static_cast<double>(z[pid]);

ph124[i] = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
ph134[i] = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];
ph144[i] = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];

ph154[i] = ph125[i] = o_x * h_ang[9] + o_y * h_ang[10] + o_z * h_ang[11];
ph164[i] = ph135[i] = o_x * h_ang[12] + o_y * h_ang[13] + o_z * h_ang[14];
ph174[i] = ph145[i] = o_x * h_ang[15] + o_y * h_ang[16] + o_z * h_ang[17];
}
}