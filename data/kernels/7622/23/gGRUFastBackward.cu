#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableLogit(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gGRUFastBackward(float* outState, float* outXW, float* outSU, float* outB, const float* state, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols, bool final) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float m = !mask || mask[j];

float* rowOutState = outState + j * cols;
float* rowOutXW = outXW + j * cols * 3;
float* rowOutSU = outSU + j * cols * 3;

const float* rowState = state + j * cols;
const float* rowXW = xW + j * cols * 3;
const float* rowSU = sU + j * cols * 3;
const float* rowAdj = adj + j * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
int k = i + cols;
int l = i + 2 * cols;

float r = stableLogit(rowXW[i] + rowSU[i] + b[i]);
float z = stableLogit(rowXW[k] + rowSU[k] + b[k]);

float h;
if(final)
h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
else
h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

float adj = rowAdj[i];

float t = (1 - z) * (1 - h * h);

// df/ds
if(outState)
rowOutState[i] += (m * z - m + 1) * adj;

// df/d(xW_r) ...
float dfdxW_r = m * r * (1 - r) * t * adj;
if(final)
dfdxW_r *= rowSU[l] + b[l];
else
dfdxW_r *= rowSU[l];
if(outXW)
rowOutXW[i] += dfdxW_r;
if(outSU)
rowOutSU[i] += dfdxW_r;
if(outB)
atomicAdd(outB + i, dfdxW_r);

// df/d(xW_z) ...
float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
if(outXW)
rowOutXW[k] += dfdxW_z;
if(outSU)
rowOutSU[k] += dfdxW_z;
if(outB)
atomicAdd(outB + k, dfdxW_z);

// df/d(xW_x) ...
float dfdxW_x = m * t * adj;
if(outXW)
rowOutXW[l] += dfdxW_x;
if(outSU)
rowOutSU[l] += dfdxW_x * r;
if(outB)
if(final)
atomicAdd(outB + l, dfdxW_x * r);
else
atomicAdd(outB + l, dfdxW_x);
}
}
}
}
}