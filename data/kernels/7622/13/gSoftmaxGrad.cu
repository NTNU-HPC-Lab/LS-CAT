#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
extern __shared__ float _share[];
float* _sum = _share + blockDim.x;

float* gradRow = grad + j * cols;
const float* adjRow = adj + j * cols;
const float* valRow = val + j * cols;
_sum[threadIdx.x] = 0.0;
for(int tid = 0; tid < cols; tid += blockDim.x) {
int id = tid + threadIdx.x;
if(id < cols) {
_sum[threadIdx.x] += valRow[id] * adjRow[id];
}
}
__syncthreads();
int len = blockDim.x;
while(len != 1) {
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
_sum[threadIdx.x] += _sum[threadIdx.x + skip];
len = (len + 1) >> 1;
}
__syncthreads();
for(int tid = 0; tid < cols; tid += blockDim.x) {
int id = tid + threadIdx.x;
if(id < cols) {
float val = valRow[id] * (adjRow[id] - _sum[0]);
if(val)
gradRow[id] += val;
}
}
}
}
}