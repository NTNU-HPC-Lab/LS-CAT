#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gCopyRows(float* out, const float* in, size_t cols, const size_t* sourceRowIdx, size_t rows) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
size_t dstId = j;
size_t srcId = sourceRowIdx[j];

float* rowOut = out + dstId * cols;
const float* rowIn = in + srcId * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols)
rowOut[i] = rowIn[i];
}
}
}
}