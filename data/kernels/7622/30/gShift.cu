#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gShift(float* out, const float* in, int length, int offset) {
for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
if(index < length) {
if(index - offset < 0 || index - offset >= length)
out[index] = 0;
else
out[index] = in[index - offset];
}
}
}