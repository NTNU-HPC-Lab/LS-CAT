#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
int n,N;
int offset,soffset;

// since we are using multiple threads per blocks as well as multiple blocks
int vidxb = 4*(blockIdx.x * blockDim.x) + threadIdx.x;
int basevert = 4*(blockIdx.x * blockDim.x);

int vidx,tab;
float4 nv,tv;
float dot,n1,n2,norm;

// create a cache for 4 elements per block (4*BLOCK_SIZE elements)
__shared__ float4 SI[4*BLOCK_SIZE_CVD];

int bidx = threadIdx.x;
// this means we have 128 neighboring vertices cached
for (vidx=vidxb; vidx<vidxb+4*BLOCK_SIZE_CVD; vidx+=BLOCK_SIZE_CVD)
{
if (vidx < nVertices)
{
SI[bidx] = V[vidx];
bidx+=BLOCK_SIZE_CVD;
}
}

__syncthreads();

bidx = threadIdx.x;
// preload the current BLOCK_SIZE vertices
for (vidx=vidxb; vidx<vidxb+4*BLOCK_SIZE_CVD; vidx+=BLOCK_SIZE_CVD)
{
if (vidx < nVertices)
{
offset = NBOFFSETS[ vidx ];
N = nNeighbors[ vidx ];
tv = SI[bidx];

bidx += BLOCK_SIZE_CVD;

for (n = 0; n < N; n++)
{
soffset = NEIGHBOR[offset+n];

/* There seems to be little to NO benefit of this local caching,
either because we have no hits, or reading from the shared memory
is just as slow as reading from global memory
*/
tab = soffset - basevert;
if (tab > 0 && tab < 4*BLOCK_SIZE_CVD)
{
nv = SI[tab];
}
else
{
nv = V[soffset];
}

// avoid FMADS
//dot = tv.x*nv.x + tv.y*nv.y + tv.z*nv.z;

dot = __fmul_rn(tv.x,nv.x);
dot = __fadd_rn(dot,__fmul_rn(tv.y,nv.y));
dot = __fadd_rn(dot,__fmul_rn(tv.z,nv.z));

//n1 = tv.x*tv.x + tv.y*tv.y + tv.z*tv.z;

n1 = __fmul_rn(tv.x,tv.x);
n1 = __fadd_rn(n1,__fmul_rn(tv.y,tv.y));
n1 = __fadd_rn(n1,__fmul_rn(tv.z,tv.z));

//n2 = nv.x*nv.x + nv.y*nv.y + nv.z*nv.z;

n2 = __fmul_rn(nv.x,nv.x);
n2 = __fadd_rn(n2,__fmul_rn(nv.y,nv.y));
n2 = __fadd_rn(n2,__fmul_rn(nv.z,nv.z));

norm = __fmul_rn(__fsqrt_rn(n1),__fsqrt_rn(n2));

//norm = __fsqrt_rn(n1) * __fsqrt_rn(n2);

// this seems to be a quell of numerical error here
if (norm < 1.0e-7f)
{
dist[offset+n] = 0.0f;
}
else if (fabsf(dot) > norm)
{
dist[offset+n] = 0.0f;
}
else
{
dist[offset+n] = __fmul_rn(circumference,fabsf(acosf(dot/norm)));
}
}
}
}
}