#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
int vidx = 4*(blockIdx.x * blockDim.x) + threadIdx.x;
int idx;
for (idx=0; idx<4*BLOCK_SIZE_AVGG; idx+=BLOCK_SIZE_AVGG)
{
D[vidx+idx] = TD[vidx+idx];
}
}