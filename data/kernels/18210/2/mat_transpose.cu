#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_transpose(const float *a, float *b, int n, int m){
const int TIlE_WIDTH = 8;
__shared__ float temp[TIlE_WIDTH][TIlE_WIDTH];

int bx = blockIdx.x, by = blockIdx.y;
int tx = threadIdx.x, ty = threadIdx.y;

int i = TIlE_WIDTH * bx + tx;
int j = TIlE_WIDTH * by + ty;
int idxa = j * n + i;
int idxb = i * n + j;

temp[ty][tx] = a[idxa];
__syncthreads();

b[idxb] = temp[ty][tx];

// if(i < n and j < m){
//     b[idxb] = a[idxa];
// }
}