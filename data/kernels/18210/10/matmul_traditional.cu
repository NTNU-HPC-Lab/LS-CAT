#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_traditional(const float *a, const float *b, float *c, int n, int m){
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;
//printf("%d %d %d %d %d %d\n",blockDim.x,blockDim.y,blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
int idx = i * n + j;


int2 i2 = make_int2(1, 2);
float4 f4 = make_float4(0, 0, 0, 0);
f4.x = 0.1, f4.y = 0.2, f4.z = 0.3, f4.w = 0.4;
//printf("%d %d %f %f %f %f\n", i2.x, i2.y, f4.x, f4.y, f4.z, f4.w);

if(i < n and j < m){
//printf("%d %d %d %d %d %d\n", i, j, idx, a[idx], b[idx], c[idx]);
float sum = 0;
for(int k = 0; k < n; k++){
int idxa = i * n + k;
int idxb = k * n + j;
sum += a[idxa] * b[idxb];
}
c[idx] = sum;
}
}