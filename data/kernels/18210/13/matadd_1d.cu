#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd_1d(const float *a, const float *b, float *c, int n, int m){
int i = blockDim.x * blockIdx.x + threadIdx.x;
//处理m个数据相加
if(i < n){
for(int j = 0; j < m; j++){
int idx = j * n + i;
c[idx] = a[idx] + b[idx];
}
}
}