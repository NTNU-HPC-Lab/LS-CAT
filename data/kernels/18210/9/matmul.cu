#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul(const float *a, const float *b, float *c, int n, int m){
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;
//printf("%d %d %d %d %d %d\n",blockDim.x,blockDim.y,blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
int idx = j * n + i;
if(i < n and j < m){
//printf("%d %d %d %d %d %d\n", i, j, idx, a[idx], b[idx], c[idx]);
float sum = 0;
for(int k = 0; k < n; k++){
int idxa = j * n + k;
int idxb = k * n + i;
sum += a[idxa] * b[idxb];
}
c[idx] = sum;
}
}