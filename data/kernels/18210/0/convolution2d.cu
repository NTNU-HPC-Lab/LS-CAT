#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define eps 1e-4

//每个thread负责output的一个pixel

__global__ void convolution2d(float *img, float *kernel, float* result, int n, int m, int kw, int kh, int out_n, int out_m, bool padding)
{
int bx = blockIdx.x, by = blockIdx.y;
int tx = threadIdx.x, ty = threadIdx.y;
int x = bx * blockDim.x + tx;
int y = by * blockDim.y + ty;
int idx = y * out_m + x;
//printf("%d %d %d %d %d %d\n", bx, by, tx, ty, x, y);
if(idx < out_n * out_m){
float ret = 0;
for(int i = 0; i < kw; i++){
for(int j = 0; j < kh; j++){
//ret += img[(y + j) * m + (x + i)] * kernel[i * kh + j];
//padding = same: (x,y) 为中心点，(x-kw/2, y-kh/2)为左上角第一个点
//padding = valid: (x+kw/2, y+kh/2)为中心点, (x,y)为左上角第一个点
int cur_x = 0, cur_y = 0;
if(padding == true){
cur_x = x - kw / 2 + i;
cur_y = y - kh / 2 + j;
}
else{
cur_x = x + i;
cur_y = y + j;
}
if(cur_x >= 0 and cur_x < n and cur_y >= 0 and cur_y < m){
ret += img[cur_y * m + cur_x] * kernel[i * kh + j];
}
}
}
//printf("%d %d %d %f\n", x, y, idx, ret);
//__syncthreads();
result[idx] = ret;
}
}