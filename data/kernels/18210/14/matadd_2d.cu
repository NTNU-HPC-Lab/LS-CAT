#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd_2d(const float *a, const float *b, float *c, int n, int m){
int i =  blockDim.x * blockIdx.x + threadIdx.x;
int j =  blockIdx.y;
if(i < n and j < m){
int idx = j * n + i;
c[idx] = a[idx] + b[idx];
}
}