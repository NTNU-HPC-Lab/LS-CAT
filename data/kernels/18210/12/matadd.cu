#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd(const float *a, const float *b, float *c, int n, int m){
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;
int idx = j * n + i;
if(i < n and j < m){
c[idx] = a[idx] + b[idx];
}
}