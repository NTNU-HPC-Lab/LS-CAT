#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_partition(const float *a, const float *b, float *c, int n){
const int TILE_WIDTH = 8;
__shared__ float na[TILE_WIDTH][TILE_WIDTH];
__shared__ float nb[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x, tx = threadIdx.x;
int by = blockIdx.y, ty = threadIdx.y;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;

float sum = 0;


//每个线程都会执行整个函数，因此每次都是不一样的(ty, tx)位置
for(int m = 0; m < n / TILE_WIDTH; m++){
na[ty][tx] = a[row * n + m * TILE_WIDTH + tx];
nb[ty][tx] = b[(ty + m * TILE_WIDTH) * n + col];
__syncthreads();
//整个tile的值都全了才能继续算

#pragma unroll TILE_WIDTH
for(int k = 0; k < TILE_WIDTH; k++){
sum += na[ty][k] * nb[k][tx];
}
__syncthreads();
//算完这一个tile才能再往里写
}
c[row * n + col] = sum;
}