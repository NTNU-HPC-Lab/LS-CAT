#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cal_hist(float *da, int *hist_da, int N, int M){
int bx = blockIdx.x;
int tx = threadIdx.x;
int idx = bx * blockDim.x + tx;
if(idx < N){
// add a lock here to make sure this (read, write) operation atomic.
atomicAdd(&hist_da[(int)da[idx]], 1);
//hist_da[(int)da[idx]] += 1;
}
}