#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulKernel(float *C, float *A, float *B, int width, int height){
int tx = blockIdx.x * blockDim.x + threadIdx.x;
int ty = blockIdx.y * blockDim.y + threadIdx.y;
if(tx >= width || ty >= height)
return;

float sum = 0;
for(int i=0; i<width; ++i){
sum += A[ty * width + i] * B[i * width + tx];
}

C[ty * width + tx] = sum;
}