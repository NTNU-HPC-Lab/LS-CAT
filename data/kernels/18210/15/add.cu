#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(const float *a, const float *b, float *c, int n){
int i = blockDim.x * blockIdx.x + threadIdx.x;
if(i < n){
c[i] = a[i] + b[i];
}
}