#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate(float *da, float* ans_device, int N){
int bx = blockIdx.x;
int tx = threadIdx.x;
int idx = bx * blockDim.x + tx;
//printf("%d\n", idx);
for(int stride = N / 2; stride > 0; stride >>= 1){
if(idx < stride){
da[idx] = da[idx] + da[idx + stride];
}
__syncthreads();
}
if(idx == 0){
ans_device[0] = da[idx];
//printf("ans 0: %f\n", ans_device[0]);
}
}