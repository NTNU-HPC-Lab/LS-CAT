#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void showgrid(){
printf("thread: %d, %d %d\nblock Idxs: %d, %d %d\nblock Dims: %d, %d %d\ngrid: %d, %d %d\n\n\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}