#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float devIoU(const float *a, const float *b){
//a: [5, ] b: [5, ], ymin, xmin, ymax, xmax, score
float w = max(0.0, min(a[2], b[2]) - max(a[0], b[0]));
float h = max(0.0, min(a[3], b[3]) - max(a[1], b[1]));
float intersect = w * h;
float sa = (a[2] - a[0]) * (a[3] - a[1]);
float sb = (b[2] - b[0]) * (b[3] - b[1]);
float _union = sa + sb - intersect;
float eps = 1e-4;
return intersect * 1.0 / (_union + eps);
}
__global__ void nms_kernel(float *bbox_dev, unsigned long long *mask_dev, int num_boxes, int col_blocks, float threshold){
//for each block(c, r) with thread(t, 0), compute the cur_box: r * 64 + t with boxes[c*64 to c*64+63], store to mask_dev
//bx = c, by = r, t = tx
int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;

//因为划block时取整，最后一组可能不满, 实际上的row对应block上面的y方向
const int row_size = min(num_boxes - by * THREADS, THREADS);
const int col_size = min(num_boxes - bx * THREADS, THREADS);

__shared__ float sh[THREADS * 5];
//put [c*64 ~ c*64+63] to share mem, i.e., in parallel: c * 64 + bx, 放入的时候可以并行放
if(tx < col_size){
int cols = tx + bx * THREADS;
#pragma unroll 5
for(int j = 0; j < 5; j++){
sh[tx * 5 + j] = bbox_dev[cols * 5 + j];
}
__syncthreads();
}

//compute cur_box at each row: r * 64 + t with shared mem
if(tx < row_size){
//compute cur with share mem
const int cur_box_idx = (by * THREADS) + tx;
float *cur_box = bbox_dev + cur_box_idx * 5;

int start = 0;
if(bx == by){
start = tx + 1;
}

unsigned long long t = 0;
for(int i = start; i < col_size; i++){
if(devIoU(cur_box, sh + tx * 5) >= threshold){
t |= (1ULL<<tx);
}
}

const int mask_idx = cur_box_idx * col_blocks + bx;
mask_dev[mask_idx] = t;
}
}