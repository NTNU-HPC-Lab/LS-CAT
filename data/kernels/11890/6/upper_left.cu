#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void upper_left(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
int idx, r, c;

idx = blockIdx.x*blockDim.x+threadIdx.x;

if( idx >= i) return;

r = i - idx;
c = i + 1 - r;

int base = r*max_cols+c;
input_itemsets[base]
= maximum( input_itemsets[base-max_cols-1]+ reference[base],
input_itemsets[base-1] - penalty,
input_itemsets[base-max_cols] - penalty);
}