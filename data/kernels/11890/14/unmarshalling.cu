#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unmarshalling(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
int i, j;

i = blockIdx.y*blockDim.y+threadIdx.y;
j = blockIdx.x*blockDim.x+threadIdx.x;

if( i >= max_rows || j >= max_cols) return;

if( (i+j) < max_rows) {
input_itemsets[i*max_cols+j] = tmp[(i+j)*max_cols+j];
}
else {
input_itemsets[i*max_cols+j] = tmp[(i+j)*max_cols+j-(i+j-max_rows+1)];
}

}