#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void middle(int *input_itemsets, int *reference, int *tmp, int max_rows, int max_cols, int i, int penalty)
{

int r, c;

r = max_rows;
c = blockIdx.x*blockDim.x+threadIdx.x;

if( c >= (max_cols - 1)) return;

tmp[r*max_cols+c] = maximum( tmp[(r-2)*max_cols+c]   + reference[(r-1-c)*max_cols+c+1],
tmp[(r-1)*max_cols+c]   - penalty,
tmp[(r-1)*max_cols+c+1] - penalty);
}