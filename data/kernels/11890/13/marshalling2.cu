#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void marshalling2(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
int i, j;

i = blockIdx.y*blockDim.y+threadIdx.y+max_rows;
j = blockIdx.x*blockDim.x+threadIdx.x;

if( i >= max_rows*2-1 || j >= max_cols) return;

if( j < max_cols-(i-max_rows+1)) {
tmp[i*max_cols+j] = input_itemsets[(max_rows-1-j)*max_cols+j+1+(i-max_rows)];
}
else {
tmp[i*max_cols+j] = 0;
}
}