#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy(int *dst, int *input_itemsets, int max_rows, int max_cols, int lb0, int lb1, int ub0, int ub1)
{
int r, c;

r = blockIdx.y*blockDim.y+threadIdx.y+lb0;
c = blockIdx.x*blockDim.x+threadIdx.x+lb1;

if( r >= ub0 || c >= ub1) return;

int idx = r*max_cols+c;
dst[idx] = input_itemsets[idx];
}