#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void needle_cuda_noshr_2( int* reference, int* matrix_cuda, int cols, int penalty, int i, int block_width)
{

int bx = blockIdx.x;
int tx = threadIdx.x;

int b_index_x = bx + block_width - i;
int b_index_y = block_width - bx -1;

int index    = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols + 1 );

for( int m = 0 ; m < BLOCK_SIZE ; m++) {
if ( tx <= m ){
int t_index_x = tx;
int t_index_y = m - tx;
int idx = index + t_index_y * cols + t_index_x;
matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
matrix_cuda[idx - 1]    - penalty,
matrix_cuda[idx - cols] - penalty);
}
}

for( int m = BLOCK_SIZE - 2 ; m >=0 ; m--) {
if ( tx <= m){
int t_index_x =  tx + BLOCK_SIZE - m -1;
int t_index_y =  BLOCK_SIZE - tx - 1;
int idx = index + t_index_y * cols + t_index_x;
matrix_cuda[idx] = maximum( matrix_cuda[idx-cols-1] + reference[idx],
matrix_cuda[idx - 1]    - penalty,
matrix_cuda[idx - cols] - penalty);
}
}
}