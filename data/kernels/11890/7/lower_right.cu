#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void lower_right(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
int idx, r, c;

idx = blockIdx.x*blockDim.x+threadIdx.x;

if( idx >= i) return;

r = max_rows-1-idx;
c = max_cols-i+idx;


int base = r*max_cols+c;
input_itemsets[base]
= maximum( input_itemsets[base-max_cols-1]+ reference[base],
input_itemsets[base-1] - penalty,
input_itemsets[base-max_cols] - penalty);
}