#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void marshalling1(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
int i, j;

i = blockIdx.y*blockDim.y+threadIdx.y;
j = blockIdx.x*blockDim.x+threadIdx.x;

if( i >= max_rows || j >= max_cols) return;

if( j <= i) {
tmp[i*max_cols+j] = input_itemsets[(i-j)*max_cols+j];
}
else {
tmp[i*max_cols+j] = 0;
}
}