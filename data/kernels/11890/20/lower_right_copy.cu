#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void lower_right_copy(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
int r, c;

r = blockIdx.y*blockDim.y+threadIdx.y+i+1;
c = blockIdx.x*blockDim.x+threadIdx.x+i+1;

if( r >= max_rows || c >= max_cols) return;

if( r == (max_cols - c + i)) {
dst[r*max_cols+c]
= maximum( input_itemsets[(r-1)*max_cols+c-1]+ reference[r*max_cols+c],
input_itemsets[r*max_cols+c-1] - penalty,
input_itemsets[(r-1)*max_cols+c] - penalty);
}
else {
dst[r*max_cols+c] = input_itemsets[r*max_cols+c];
}
}