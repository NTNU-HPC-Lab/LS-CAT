#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void upper_left(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty)
{
int r, c;

r = blockIdx.y*blockDim.y+threadIdx.y+1;
c = blockIdx.x*blockDim.x+threadIdx.x+1;

if( r >= i+1 || c >= i+1) return;

if( r == (i - c + 1)) {
int base = r*max_cols+c;
dst[base] = maximum( input_itemsets[base-max_cols-1]+ reference[base],
input_itemsets[base-1] - penalty,
input_itemsets[base-max_cols] - penalty);
}
}