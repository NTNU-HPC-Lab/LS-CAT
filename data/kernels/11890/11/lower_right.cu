#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void lower_right(int *input_itemsets, int *reference, int *tmp, int max_rows, int max_cols, int i, int penalty)
{

int r, c;

r = i;
c = blockIdx.x*blockDim.x+threadIdx.x;

if( c >= (max_cols-(i-max_rows+1))) return;

tmp[r*max_cols+c] = maximum( tmp[(r-2)*max_cols+c+1]+ reference[(max_rows-1-c)*max_cols+c+(i-max_rows+1)],
tmp[(r-1)*max_cols+c] - penalty,
tmp[(r-1)*max_cols+c+1] - penalty);
}