#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum( int a, int b, int c){
int k;
if( a <= b )
k = b;
else
k = a;

if( k <=c )
return(c);
else
return(k);
}
__global__ void upper_left(int *input_itemsets, int *reference, int *tmp, int max_rows, int max_cols, int i, int penalty)
{



int r, c;

r = i;
c = blockIdx.x*blockDim.x+threadIdx.x+1;

if( c >= i) return;

tmp[r*max_cols+c] = maximum( tmp[(r-2)*max_cols+c-1] + reference[(r-c)*max_cols+c],
tmp[(r-1)*max_cols+c-1] - penalty,
tmp[(r-1)*max_cols+c]   - penalty);
}