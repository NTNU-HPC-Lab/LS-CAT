#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_density_rg_1_real_gpu_kernel(int size__, double const* psi_rg__, double wt__, double* density_rg__)
{
int ir = blockIdx.x * blockDim.x + threadIdx.x;
if (ir < size__) {
double p = psi_rg__[ir];
density_rg__[ir] += p * p * wt__;
}
}