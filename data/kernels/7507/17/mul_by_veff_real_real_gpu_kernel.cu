#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_by_veff_real_real_gpu_kernel(int nr__, double* buf__, double const* veff__)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i < nr__) {
buf__[i] *= veff__[i];
}
}