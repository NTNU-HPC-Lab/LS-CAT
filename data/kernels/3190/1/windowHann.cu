#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowHann(float* idata, int length)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
if (tidx < length)
{
idata[tidx] = 0.5*(1 + cos(2*tidx*PI_F / (length - 1)));
}
}