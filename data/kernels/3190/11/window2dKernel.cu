#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void window2dKernel(float* idata, float* window, int width, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
if(tidx < width && tidy < height)
{
idata[tidy * width + tidx] = window[tidy * width + tidx] * idata[tidy * width + tidx];
}
}