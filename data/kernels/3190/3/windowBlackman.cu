#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBlackman(float* idata, int length)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
if (tidx < length)
{
idata[tidx] = 0.74 / 2 * -0.5 * cos(2 * PI_F*tidx / (length - 1)) + 0.16 / 2 * sin(4 * PI_F*tidx / (length - 1));
}
}