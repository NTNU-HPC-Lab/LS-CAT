#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zeroFillingKernel(float* idata, int row, int length, int height)
{
int tidx = blockIdx.x * blockDim.x + threadIdx.x;
int tidy = blockIdx.y * blockDim.y;
if(tidx < length &&  tidy < height)
{
//printf("idata[%d][%d]: = %f\n", (row+tidy), tidx,idata[tidx + (row+tidy) *length]);
idata[tidx + (row+tidy) *length] = 0;
idata[tidx + (row-tidy) *length] = 0;
//printf("idata[%d][%d]: = %f\n", (row+tidy), tidx,idata[tidx + (row+tidy) *length]);

}
}