#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowKernel(float* idata, float* window, int width, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
if(tidx < width && tidy < height)
{
idata[tidy * width + tidx] = window[tidx] * idata[tidy * width + tidx];
}
}