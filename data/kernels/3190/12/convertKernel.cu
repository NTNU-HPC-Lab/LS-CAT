#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertKernel(short* idata, float* odata, int size)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
if(tidx < size)
odata[tidx] = (float)idata[tidx];
}