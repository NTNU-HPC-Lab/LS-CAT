#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowHann2d(float* idata, int length, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
if (tidx < length && tidy < height)
{
idata[tidy * length + tidx] =  0.5*(1 + cos(2*tidy*PI_F / (height - 1))) * 0.5*(1 + cos(2*tidx*PI_F / (length - 1)));
}
}