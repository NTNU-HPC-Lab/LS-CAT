#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBartlett(float* idata, int length)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
if (tidx < length)
{
idata[tidx] = 0;
}
}