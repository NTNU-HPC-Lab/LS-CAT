#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowHamming(float* idata, int length)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
if (tidx < length)
{
idata[tidx] = 0.54 - 0.46 * cos(2*tidx*PI_F / (length - 1));
}
}