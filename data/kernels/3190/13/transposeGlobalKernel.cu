#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeGlobalKernel(float* idata, float* odata, int width, int height)
{
int tidx = blockIdx.x * blockDim.x + threadIdx.x;
int tidy = blockIdx.y * blockDim.y+ threadIdx.y;

if(tidx < width && tidy < height)
{
odata[tidx*height + tidy] = idata[tidy*width + tidx];
}
}