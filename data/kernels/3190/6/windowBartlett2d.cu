#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBartlett2d(float* idata, int length, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
if (tidx < length && tidy < height)
{
idata[tidy * length + tidx] = 0;
}
}