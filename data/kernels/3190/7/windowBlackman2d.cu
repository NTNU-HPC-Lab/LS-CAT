#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBlackman2d(float* idata, int length, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
if (tidx < length && tidy < height)
{
idata[tidy * length + tidx] = (0.74 / 2 * -0.5 * cos(2 * PI_F*tidy / (height - 1)) + 0.16 / 2 * sin(4 * PI_F*tidy / (height - 1)))
* (0.74 / 2 * -0.5 * cos(2 * PI_F*tidx / (length - 1)) + 0.16 / 2 * sin(4 * PI_F*tidx / (length - 1)));
}
}