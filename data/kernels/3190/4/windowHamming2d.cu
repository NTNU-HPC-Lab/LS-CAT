#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowHamming2d(float* idata, int length, int height)
{
int tidx = threadIdx.x + blockIdx.x*blockDim.x;
int tidy = threadIdx.y + blockIdx.y*blockDim.y;
//printf("tidy: %d, tidy:%d, idx:%d", tidy,tidx ,tidy * length + tidx);
if (tidx < length && tidy < height)
{
//printf("tidy: %d, tidy:%d, idx:%d", tidy,tidx ,tidy * length + tidx);
idata[tidy * length + tidx] = (0.54 - 0.46 * cos(2*tidy*PI_F / (height - 1))) * (0.54 - 0.46 * cos(2*tidx*PI_F / (length - 1)));
}
}