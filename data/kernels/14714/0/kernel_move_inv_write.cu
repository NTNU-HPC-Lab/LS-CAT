#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
if (ptr >= (unsigned int*) end_ptr) {
return;
}

for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i++){
ptr[i] = pattern;
}

return;
}