#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "make_pillar_feature_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *dev_pillar_x_in_coors = NULL;
hipMalloc(&dev_pillar_x_in_coors, XSIZE*YSIZE);
float *dev_pillar_y_in_coors = NULL;
hipMalloc(&dev_pillar_y_in_coors, XSIZE*YSIZE);
float *dev_pillar_z_in_coors = NULL;
hipMalloc(&dev_pillar_z_in_coors, XSIZE*YSIZE);
float *dev_pillar_i_in_coors = NULL;
hipMalloc(&dev_pillar_i_in_coors, XSIZE*YSIZE);
float *dev_pillar_x = NULL;
hipMalloc(&dev_pillar_x, XSIZE*YSIZE);
float *dev_pillar_y = NULL;
hipMalloc(&dev_pillar_y, XSIZE*YSIZE);
float *dev_pillar_z = NULL;
hipMalloc(&dev_pillar_z, XSIZE*YSIZE);
float *dev_pillar_i = NULL;
hipMalloc(&dev_pillar_i, XSIZE*YSIZE);
int *dev_x_coors = NULL;
hipMalloc(&dev_x_coors, XSIZE*YSIZE);
int *dev_y_coors = NULL;
hipMalloc(&dev_y_coors, XSIZE*YSIZE);
float *dev_num_points_per_pillar = NULL;
hipMalloc(&dev_num_points_per_pillar, XSIZE*YSIZE);
const int max_points = 1;
const int GRID_X_SIZE = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
make_pillar_feature_kernel<<<gridBlock,threadBlock>>>(dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,dev_pillar_x,dev_pillar_y,dev_pillar_z,dev_pillar_i,dev_x_coors,dev_y_coors,dev_num_points_per_pillar,max_points,GRID_X_SIZE);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
make_pillar_feature_kernel<<<gridBlock,threadBlock>>>(dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,dev_pillar_x,dev_pillar_y,dev_pillar_z,dev_pillar_i,dev_x_coors,dev_y_coors,dev_num_points_per_pillar,max_points,GRID_X_SIZE);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
make_pillar_feature_kernel<<<gridBlock,threadBlock>>>(dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,dev_pillar_x,dev_pillar_y,dev_pillar_z,dev_pillar_i,dev_x_coors,dev_y_coors,dev_num_points_per_pillar,max_points,GRID_X_SIZE);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}