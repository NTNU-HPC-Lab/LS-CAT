#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computeScoreGradientList.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *trans_x = NULL;
hipMalloc(&trans_x, XSIZE*YSIZE);
float *trans_y = NULL;
hipMalloc(&trans_y, XSIZE*YSIZE);
float *trans_z = NULL;
hipMalloc(&trans_z, XSIZE*YSIZE);
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int *starting_voxel_id = NULL;
hipMalloc(&starting_voxel_id, XSIZE*YSIZE);
int *voxel_id = NULL;
hipMalloc(&voxel_id, XSIZE*YSIZE);
int valid_points_num = 1;
double *centroid_x = NULL;
hipMalloc(&centroid_x, XSIZE*YSIZE);
double *centroid_y = NULL;
hipMalloc(&centroid_y, XSIZE*YSIZE);
double *centroid_z = NULL;
hipMalloc(&centroid_z, XSIZE*YSIZE);
int voxel_num = 1;
double *e_x_cov_x = NULL;
hipMalloc(&e_x_cov_x, XSIZE*YSIZE);
double *cov_dxd_pi = NULL;
hipMalloc(&cov_dxd_pi, XSIZE*YSIZE);
double gauss_d1 = 1;
int valid_voxel_num = 1;
double *score_gradients = NULL;
hipMalloc(&score_gradients, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeScoreGradientList<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,voxel_num,e_x_cov_x,cov_dxd_pi,gauss_d1,valid_voxel_num,score_gradients);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computeScoreGradientList<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,voxel_num,e_x_cov_x,cov_dxd_pi,gauss_d1,valid_voxel_num,score_gradients);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computeScoreGradientList<<<gridBlock,threadBlock>>>(trans_x,trans_y,trans_z,valid_points,starting_voxel_id,voxel_id,valid_points_num,centroid_x,centroid_y,centroid_z,voxel_num,e_x_cov_x,cov_dxd_pi,gauss_d1,valid_voxel_num,score_gradients);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}