#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter_kernel( int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
int i_pillar = blockIdx.x;
int i_feature = threadIdx.x;
int x_ind = x_coors[i_pillar];
int y_ind = y_coors[i_pillar];
float feature = pfe_output[i_feature*MAX_NUM_PILLARS_ + i_pillar];
scattered_feature[i_feature*GRID_Y_SIZE*GRID_X_SIZE + y_ind * GRID_X_SIZE + x_ind] = feature;
}