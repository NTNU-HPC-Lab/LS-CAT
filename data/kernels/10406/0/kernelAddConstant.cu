#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* Multi-GPU sample using OpenMP for threading on the CPU side
* needs a compiler that supports OpenMP 2.0
*/


using namespace std;

// a simple kernel that simply increments each array element by b

// a predicate that checks whether each array elemen is set to its index plus b
__global__ void kernelAddConstant(int *g_a, const int b)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
g_a[idx] += b;
}