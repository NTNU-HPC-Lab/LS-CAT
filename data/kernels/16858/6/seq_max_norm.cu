#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void seq_max_norm(float* mat1, int row, int col, float* norm){
*norm = 0;
for(int i=0; i<row; i++){
for(int j =0; j<col; j++)
*norm = max(abs(mat1[i*col+j]), *norm);
}
}