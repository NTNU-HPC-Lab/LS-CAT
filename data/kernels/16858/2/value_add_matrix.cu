#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void value_add_matrix(float* mat1, float* mat2, int row, int col, float v){
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size = row*col;
if(id<size){
mat2[id] = mat1[id] + v;
}
}