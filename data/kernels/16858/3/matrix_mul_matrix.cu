#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_mul_matrix(float *A, float *B, float *C, int col_A, int col_B, int row_C, int col_C){
float sum = 0.0f;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

if (row < row_C && col < col_C) {
for (int i = 0; i < col_A; ++i) {
sum += A[row * col_A + i] * B[i * col_B + col];
}
C[row * col_B + col] = sum;
}
}