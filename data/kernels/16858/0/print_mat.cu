#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_mat(float* mat, int row, int col){
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id==0){
for(int i=0; i<row; i++){
for(int j =0; j<col; j++)
printf("%0.3f\t", mat[i*col+j]);
printf("\n");
}
printf("\n");
}
}