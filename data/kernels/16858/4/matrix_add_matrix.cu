#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_add_matrix(float* mat1, float* mat2, float* mat3, int row, int col, int sign){
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size = row*col;
if(id<size){
mat3[id] = mat1[id] + sign*mat2[id];
}
}