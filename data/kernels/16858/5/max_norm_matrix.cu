#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_norm_matrix(float* mat1, int row, int col, int* norm, float* final_norm){
*norm = 0;
__syncthreads();
int id = blockIdx.x * blockDim.x + threadIdx.x;
int size = row*col;
if(id<size){
atomicMax(norm, __float_as_int(abs(mat1[id])));
}
__syncthreads();
if(id==0){
*final_norm = __int_as_float(*norm);
}
}