#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Implements vector
*/

#ifdef DEBUG
#endif



__global__ void kern_vec_add_(float* x, float* y, float* r, size_t dim)
{
size_t _strd = blockDim.x * gridDim.x;
for(size_t _i = blockIdx.x * blockDim.x + threadIdx.x; _i < dim; _i += _strd)
r[_i] = x[_i] + y[_i];
}