#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void adicionarKernel(double* resultado, const double* n) {
int i = threadIdx.x;
double a = 1, b = 0;
double delta = pow(b, 2) - (4 * a * (n[i] * -1));
resultado[i] = ((b * -1) + sqrt(delta)) / 2 * a;
}