#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tile_kernel(const float* in,float* out, int num_planes, int num_rows, int num_cols) {

const int gid = threadIdx.x + blockIdx.x * blockDim.x;
const int elems_per_plane = num_rows * num_cols;

const int plane = gid / num_rows;
const int row   = gid % num_rows;

if (plane >= num_planes)
return;

for (int col=0;col<num_cols; ++col){
out[plane * elems_per_plane + row * num_cols + col]=in[plane*num_cols+col];
}
}