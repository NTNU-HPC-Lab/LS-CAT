#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warp_kernel(float* out, const float* in, const int* index,const float* weights,const int npixels,const int nchannels){
int pixel   = blockIdx.x * blockDim.x + threadIdx.x;
int channel = blockIdx.y * blockDim.y + threadIdx.y;
if( channel >= nchannels||pixel >= npixels)
return;
out[nchannels*pixel+channel]=in[nchannels*index[4*pixel]+channel]*weights[4*pixel]
+in[nchannels*index[4*pixel+1]+channel]*weights[4*pixel+1]
+in[nchannels*index[4*pixel+2]+channel]*weights[4*pixel+2]
+in[nchannels*index[4*pixel+3]+channel]*weights[4*pixel+3];
}