#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ColumnReduceSimpleKernel(const float* in,float* out, int num_planes, int num_rows, int num_cols) {

const int gid = threadIdx.x + blockIdx.x * blockDim.x;
const int elems_per_plane = num_rows * num_cols;

const int plane = gid / num_cols;
const int col = gid % num_cols;

if (plane >= num_planes)
return;

float sum = in[plane * elems_per_plane + col]+in[plane * elems_per_plane + num_cols + col];
for (int row = 2; row < num_rows; ++row) {
sum = sum+in[plane * elems_per_plane + row * num_cols + col];
}
out[plane * num_cols + col] = sum;
}