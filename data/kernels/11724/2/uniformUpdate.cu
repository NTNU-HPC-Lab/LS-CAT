#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/



//All three kernels run 512 threads per workgroup
//Must be a power of two
#define THREADBLOCK_SIZE 1024

////////////////////////////////////////////////////////////////////////////////
// Basic scan codelets
////////////////////////////////////////////////////////////////////////////////
//Naive inclusive scan: O(N * log2(N)) operations
//Allocate 2 * 'size' local memory, initialize the first half
//with 'size' zeros avoiding if(pos >= offset) condition evaluation
//and saving instructions
__global__ void uniformUpdate( uint4 *d_Data, uint *d_Buffer )
{
__shared__ uint buf;
uint pos = blockIdx.x * blockDim.x + threadIdx.x;

if (threadIdx.x == 0)
{
buf = d_Buffer[blockIdx.x];
}

__syncthreads();

uint4 data4 = d_Data[pos];
data4.x += buf;
data4.y += buf;
data4.z += buf;
data4.w += buf;
d_Data[pos] = data4;
}