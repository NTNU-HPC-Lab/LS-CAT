#include "hip/hip_runtime.h"
#include "includes.h"

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 4)
*  -> Features: none (basic tiled version, using only global memory)
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*     - coalesced memory access
*     - overlapping loads of subsequent tile pairs (using registers & shared memory)
*/

__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n)
{
__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

int row = blockIdx.y*TILE_SIZE + threadIdx.y;
int column = blockIdx.x*TILE_SIZE + threadIdx.x;

if(row >=n || column >=n)
{
return;
}

float Celem = 0.0;
for(int m = 0; m < n/TILE_SIZE;m++)
{
*(*(A_shared + threadIdx.y) + threadIdx.x) = *(Ad + row*n + m*TILE_SIZE + threadIdx.x);
*(*(B_shared + threadIdx.y) + threadIdx.x) = *(Bd + (m*TILE_SIZE + threadIdx.y)*n + column);
__syncthreads();

for(int k  = 0;k<TILE_SIZE;k++)
{
Celem += *(*(A_shared + threadIdx.y) + k)*(*(*(B_shared + k) + threadIdx.x));
}
__syncthreads();

}
*(Cd +  row*n + column) = Celem;

}