#include "hip/hip_runtime.h"
#include "includes.h"

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 4)
*  -> Features: none (basic tiled version, using only global memory)
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*     - coalesced memory access
*     - overlapping loads of subsequent tile pairs (using registers & shared memory)
*/

__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n)
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int column = blockIdx.x*blockDim.x + threadIdx.x;

if(row >=n || column >=n)
{
return;
}

float Celem = 0.0;
for(int j = 0;j<n;j++)
{
Celem += *(Ad + row*n + j)*(*(Bd + j*n + column));
}

*(Cd + row*n + column) = Celem;

}