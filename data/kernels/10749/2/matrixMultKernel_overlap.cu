#include "hip/hip_runtime.h"
#include "includes.h"

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 4)
*  -> Features: none (basic tiled version, using only global memory)
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*/

/*
* matrix multiplication C += A*B
*  -> CUDA kernel
*     (implementation adopted from Kirk&Hwu:
*      "Programming Massively Parallel Processors, chapter 5)
*  -> Features:
*     - tiled matrix multiplication with use of shared memory
*     - coalesced memory access
*     - overlapping loads of subsequent tile pairs (using registers & shared memory)
*/

__global__ void matrixMultKernel_overlap(float* Ad, float* Bd, float* Cd, int n)

{
__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

int row = blockIdx.y*TILE_SIZE + threadIdx.y;
int column = blockIdx.x*TILE_SIZE + threadIdx.x;

if(row >= n || column >=n)
{
return;
}

float Celem = 0.0;
float reg_1 = *(Ad + row*n + threadIdx.x);
float reg_2 = *(Bd + threadIdx.y*n + column);

for(int m = 1;m<n/TILE_SIZE;m++)
{
A_shared[threadIdx.y][threadIdx.x] = reg_1;
B_shared[threadIdx.y][threadIdx.x] = reg_2;

__syncthreads();

reg_1 = *(Ad + row*n + m*TILE_SIZE + threadIdx.x);
reg_2 = *(Bd + (m*TILE_SIZE + threadIdx.y)*n + column);

for(int k = 0;k<TILE_SIZE;k++)
{
Celem += A_shared[threadIdx.y][k]*B_shared[k][threadIdx.x];
}

__syncthreads();
}

A_shared[threadIdx.y][threadIdx.x] = reg_1;
B_shared[threadIdx.y][threadIdx.x] = reg_2;

__syncthreads();

for(int k = 0;k<TILE_SIZE;k++)
{
Celem += A_shared[threadIdx.y][k]*B_shared[k][threadIdx.x];
}

__syncthreads();

*(Cd + row*n + column) = Celem;

}