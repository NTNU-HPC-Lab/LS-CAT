#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float gamma_correction(float f_stop, float gamma, float val)
{
return powf((val*powf(2.0,f_stop)),(1.0/gamma));
}
__device__ float rgb2Lum(float B, float G, float R)
{
return B * 0.0722 + G * 0.7152 + R * 0.2126;
}
__global__ void gamma_tonemap_kernel(float* imageIn, float* imageOut, int width, int height, int channels, float f_stop, float gamma)
{
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

if(Row < height && Col < width) {
float B, G, R, L, nL, scale;
B = imageIn[(Row*width+Col)*3+BLUE];
G = imageIn[(Row*width+Col)*3+GREEN];
R = imageIn[(Row*width+Col)*3+RED];

L = rgb2Lum(B, G, R);
nL = gamma_correction(f_stop, gamma, L);
scale = nL / L;

imageOut[(Row*width+Col)*3+BLUE] = B * scale;
imageOut[(Row*width+Col)*3+GREEN] = G * scale;
imageOut[(Row*width+Col)*3+RED] = R * scale;
}
}