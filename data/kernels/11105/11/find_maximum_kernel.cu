#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float rgb2Lum(float B, float G, float R)
{
return B * 0.0722 + G * 0.7152 + R * 0.2126;
}
__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
unsigned int stride = gridDim.x*blockDim.x;
unsigned int offset = 0;
//	const int size = blockSize;

extern __shared__ float cache[];

float temp = -1.0;
while(((index + offset)*3 + 2) < n) {
float B, G, R, L;

B = array[(index + offset)*3 + BLUE];
G = array[(index + offset)*3 + GREEN];
R = array[(index + offset)*3 + RED];
L = rgb2Lum(B, G, R);

temp = fmaxf(temp, L);

offset += stride;
}

cache[threadIdx.x] = temp;

__syncthreads();


// reduction
unsigned int i = blockDim.x/2;
while(i != 0){
if(threadIdx.x < i){
cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
}

__syncthreads();
i /= 2;
}

if(threadIdx.x == 0){
while(atomicCAS(mutex,0,1) != 0);  //lock
*max = fmaxf(*max, cache[0]);
atomicExch(mutex, 0);  //unlock
}
}