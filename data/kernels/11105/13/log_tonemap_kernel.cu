#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float logarithmic_mapping(float k, float q, float val_pixel, float maxLum)
{
return (log10f(1.0 + q * val_pixel))/(log10f(1.0 + k * maxLum));
}
__device__ float rgb2Lum(float B, float G, float R)
{
return B * 0.0722 + G * 0.7152 + R * 0.2126;
}
__global__ void log_tonemap_kernel(float* imageIn, float* imageOut, int width, int height, int channels, float k, float q, float* max)
{
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

if(Row < height && Col < width) {
float B, G, R, L, nL, scale;
B = imageIn[(Row*width+Col)*3+BLUE];
G = imageIn[(Row*width+Col)*3+GREEN];
R = imageIn[(Row*width+Col)*3+RED];

L = rgb2Lum(B, G, R);
nL = logarithmic_mapping(k, q, L, *max);
scale = nL / L;

imageOut[(Row*width+Col)*3+BLUE] = B * scale;
imageOut[(Row*width+Col)*3+GREEN] = G * scale;
imageOut[(Row*width+Col)*3+RED] = R * scale;
}
}