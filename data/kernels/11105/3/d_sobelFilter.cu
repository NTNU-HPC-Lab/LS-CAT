#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_sobelFilter(unsigned char* imageIn, unsigned char* imageOut, int width, int height, int maskWidth, char* M) {
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;

int nrow = Row - (maskWidth/2);
int ncol = Col - (maskWidth/2);
int res = 0;

if(Row < height && Col < width) {
for(int i=0; i<maskWidth; i++) {
for(int j=0; j<maskWidth; j++) {
if((nrow + i >= 0 && nrow + i < height) && (ncol + j >= 0 && ncol + j < width)) {
res += imageIn[(nrow + i)*width + (ncol + j)] * M[i*maskWidth + j];
}
}
}
if(res < 0)
res = 0;
else
if(res > 255)
res = 255;
imageOut[Row*width+Col] = (unsigned char)res;
}
}