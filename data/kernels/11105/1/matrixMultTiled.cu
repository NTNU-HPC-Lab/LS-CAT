#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultTiled(float* d_A, float* d_B, float* d_C, int width) {
__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

// Identify the row and column of the d_P element to work on
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;

float ans = 0;
// Loop over the d_M and d_N tiles required to compute d_P element
for(int m=0; m < (TILE_WIDTH + width - 1)/TILE_WIDTH; m++) {
// Coolaborative loading of d_M and d_N tiles into shared memory
if(m*TILE_WIDTH + tx < width && Row < width)
ds_A[ty][tx] = d_A[Row*width + m*TILE_WIDTH + tx];
else
ds_A[ty][tx] = 0.0;
if(m*TILE_WIDTH + ty < width && Col < width)
ds_B[ty][tx] = d_B[(m*TILE_WIDTH + ty) * width + Col];
else
ds_B[ty][tx] = 0.0;

__syncthreads();

for(int k=0; k<TILE_WIDTH; k++) {
ans += ds_A[ty][k] * ds_B[k][tx];
}
__syncthreads();
}
if(Row < width && Col < width)
d_C[Row*width+Col]=ans;
}