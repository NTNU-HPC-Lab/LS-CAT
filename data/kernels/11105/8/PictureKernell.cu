#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PictureKernell(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m) {
int Row = blockIdx.y*blockDim.y + threadIdx.y;
int Col = blockIdx.x*blockDim.x + threadIdx.x;

if((Row < m) && (Col < n)) {
d_Pout[Row*n+Col] = 2*d_Pin[Row*n+Col];
}
}