#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* d_A, float* d_B, float* d_C) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
if(i<TAM)
d_C[i] = d_A[i] + d_B[i];
}