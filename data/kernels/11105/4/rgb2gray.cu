#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2gray(unsigned char* d_Pin, unsigned char* d_Pout, int width, int height) {
int Row = blockIdx.y*blockDim.y + threadIdx.y;
int Col = blockIdx.x*blockDim.x + threadIdx.x;

if((Row < height) && (Col < width)) {
d_Pout[Row*width+Col] = d_Pin[(Row*width+Col)*3+BLUE]*0.114 + d_Pin[(Row*width+Col)*3+GREEN]*0.587 + d_Pin[(Row*width+Col)*3+RED]*0.299;

}
}