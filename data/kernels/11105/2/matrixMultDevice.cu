#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultDevice(float* d_A, float* d_B, float* d_C, int width) {
int Row = blockDim.y * blockIdx.y + threadIdx.y;
int Col = blockDim.x * blockIdx.x + threadIdx.x;
if(Row < width && Col < width) {
float ans = 0.0;
for(int k=0; k<width; k++) {
ans += d_A[Row*width+k]*d_B[k*width+Col];
}
d_C[Row*width+Col]=ans;
}
}