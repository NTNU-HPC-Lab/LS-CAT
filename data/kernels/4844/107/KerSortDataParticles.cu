#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n,unsigned pini,const unsigned *sortpart,const double2 *a,const double *b,const float4 *c,double2 *a2,double *b2,float4 *c2)
{
const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Particle number.
if(p<n){
const unsigned oldpos=(p<pini? p: sortpart[p]);
a2[p]=a[oldpos];
b2[p]=b[oldpos];
c2[p]=c[oldpos];
}
}