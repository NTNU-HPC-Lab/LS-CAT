#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "KerFtCalcForcesRes.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
unsigned ftcount = 1;
bool simulate2d = 1;
double dt = 1;
const float3 *ftoomega = NULL;
hipMalloc(&ftoomega, XSIZE*YSIZE);
const float3 *ftovel = NULL;
hipMalloc(&ftovel, XSIZE*YSIZE);
const double3 *ftocenter = NULL;
hipMalloc(&ftocenter, XSIZE*YSIZE);
const float3 *ftoforces = NULL;
hipMalloc(&ftoforces, XSIZE*YSIZE);
float3 *ftoforcesres = NULL;
hipMalloc(&ftoforcesres, XSIZE*YSIZE);
double3 *ftocenterres = NULL;
hipMalloc(&ftocenterres, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
KerFtCalcForcesRes<<<gridBlock,threadBlock>>>(ftcount,simulate2d,dt,ftoomega,ftovel,ftocenter,ftoforces,ftoforcesres,ftocenterres);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
KerFtCalcForcesRes<<<gridBlock,threadBlock>>>(ftcount,simulate2d,dt,ftoomega,ftovel,ftocenter,ftoforces,ftoforcesres,ftocenterres);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
KerFtCalcForcesRes<<<gridBlock,threadBlock>>>(ftcount,simulate2d,dt,ftoomega,ftovel,ftocenter,ftoforces,ftoforcesres,ftocenterres);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}