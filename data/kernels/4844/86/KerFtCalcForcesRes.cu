#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerFtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt ,const float3 *ftoomega,const float3 *ftovel,const double3 *ftocenter,const float3 *ftoforces ,float3 *ftoforcesres,double3 *ftocenterres)
{
const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
if(cf<ftcount){
//-Compute fomega.
float3 fomega=ftoomega[cf];
{
const float3 omegaace=ftoforces[cf*2+1];
fomega.x=float(dt*omegaace.x+fomega.x);
fomega.y=float(dt*omegaace.y+fomega.y);
fomega.z=float(dt*omegaace.z+fomega.z);
}
float3 fvel=ftovel[cf];
//-Zero components for 2-D simulation. | Anula componentes para 2D.
float3 face=ftoforces[cf*2];
if(simulate2d){ face.y=0; fomega.x=0; fomega.z=0; fvel.y=0; }
//-Compute fcenter.
double3 fcenter=ftocenter[cf];
fcenter.x+=dt*fvel.x;
fcenter.y+=dt*fvel.y;
fcenter.z+=dt*fvel.z;
//-Compute fvel.
fvel.x=float(dt*face.x+fvel.x);
fvel.y=float(dt*face.y+fvel.y);
fvel.z=float(dt*face.z+fvel.z);
//-Store data to update floating. | Guarda datos para actualizar floatings.
ftoforcesres[cf*2]=fomega;
ftoforcesres[cf*2+1]=fvel;
ftocenterres[cf]=fcenter;
}
}