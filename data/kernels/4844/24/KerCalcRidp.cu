#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const unsigned *idp,unsigned *ridp)
{
unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(p<n){
p+=ini;
const unsigned id=idp[p];
if(idini<=id && id<idfin)ridp[id-idini]=p;
}
}