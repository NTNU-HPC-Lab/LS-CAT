#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerComputeSpsTau(unsigned n,unsigned pini,float smag,float blin ,const float4 *velrhop,const float2 *gradvelff,float2 *tauff)
{
unsigned p=blockIdx.x*blockDim.x + threadIdx.x;
if(p<n){
const unsigned p1=p+pini;
float2 rr=gradvelff[p1*3];   const float grad_xx=rr.x,grad_xy=rr.y;
rr=gradvelff[p1*3+1]; const float grad_xz=rr.x,grad_yy=rr.y;
rr=gradvelff[p1*3+2]; const float grad_yz=rr.x,grad_zz=rr.y;
const float pow1=grad_xx*grad_xx + grad_yy*grad_yy + grad_zz*grad_zz;
const float prr= grad_xy*grad_xy + grad_xz*grad_xz + grad_yz*grad_yz + pow1+pow1;
const float visc_sps=smag*sqrt(prr);
const float div_u=grad_xx+grad_yy+grad_zz;
const float sps_k=(2.0f/3.0f)*visc_sps*div_u;
const float sps_blin=blin*prr;
const float sumsps=-(sps_k+sps_blin);
const float twovisc_sps=(visc_sps+visc_sps);
float one_rho2=1.0f/velrhop[p1].w;
//-Computes new values of tau[].
const float tau_xx=one_rho2*(twovisc_sps*grad_xx +sumsps);
const float tau_xy=one_rho2*(visc_sps   *grad_xy);
tauff[p1*3]=make_float2(tau_xx,tau_xy);
const float tau_xz=one_rho2*(visc_sps   *grad_xz);
const float tau_yy=one_rho2*(twovisc_sps*grad_yy +sumsps);
tauff[p1*3+1]=make_float2(tau_xz,tau_yy);
const float tau_yz=one_rho2*(visc_sps   *grad_yz);
const float tau_zz=one_rho2*(twovisc_sps*grad_zz +sumsps);
tauff[p1*3+2]=make_float2(tau_yz,tau_zz);
}
}