#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n,unsigned pini,const unsigned *sortpart,const float *a,const float *b,float *a2,float *b2)
{
const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Particle number.
if(p<n){
const unsigned oldpos=(p<pini? p: sortpart[p]);
a2[p]=a[oldpos];
b2[p]=b[oldpos];
}
}