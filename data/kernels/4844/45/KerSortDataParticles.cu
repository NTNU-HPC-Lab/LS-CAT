#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n,unsigned pini,const unsigned *sortpart,const float4 *a,float4 *a2)
{
const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Particle number.
if(p<n){
const unsigned oldpos=(p<pini? p: sortpart[p]);
a2[p]=a[oldpos];
}
}