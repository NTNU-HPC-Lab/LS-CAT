#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerComputeVelMod(unsigned n,const float4 *vel,float *velmod)
{
unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(p<n){
const float4 r=vel[p];
velmod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
}
}