#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerComputeAceMod(unsigned n,const float3 *ace,float *acemod)
{
unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(p<n){
const float3 r=ace[p];
acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
}
}