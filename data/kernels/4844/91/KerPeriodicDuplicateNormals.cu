#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerPeriodicDuplicateNormals(unsigned n,unsigned pini,const unsigned *listp,float3 *normals,float3 *motionvel)
{
const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(p<n){
const unsigned pnew=p+pini;
const unsigned rp=listp[p];
const unsigned pcopy=(rp&0x7FFFFFFF);
normals[pnew]=normals[pcopy];
if(motionvel)motionvel[pnew]=motionvel[pcopy];
}
}