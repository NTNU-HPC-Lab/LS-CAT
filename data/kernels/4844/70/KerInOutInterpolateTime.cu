#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerInOutInterpolateTime(unsigned npt,double fxtime ,const float *vel0,const float *vel1,float *vel)
{
const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(p<npt){
const float v0=vel0[p];
vel[p]=float(fxtime*(vel1[p]-v0)+v0);
}
}