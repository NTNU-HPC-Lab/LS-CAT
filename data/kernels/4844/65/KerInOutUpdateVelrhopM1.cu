#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerInOutUpdateVelrhopM1(unsigned n,const int *inoutpart ,const float4 *velrhop,float4 *velrhopm1)
{
const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
if(cp<n){
const unsigned p=inoutpart[cp];
velrhopm1[p]=velrhop[p];
}
}