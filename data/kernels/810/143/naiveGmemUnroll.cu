#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naiveGmemUnroll(float *out, float *in, const int nx, const int ny)
{
unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

unsigned int ti = iy * nx + ix;
unsigned int to = ix * ny + iy;

if (ix + blockDim.x < nx && iy < ny)
{
out[to]                   = in[ti];
out[to + ny * blockDim.x]   = in[ti + blockDim.x];
}
}