#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyWithUVAOffset(float *A, float *B, float *C, const int N, int offset)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < N) C[i + offset] = A[i + offset] + B[i + offset];
}