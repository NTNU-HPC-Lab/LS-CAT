#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;

// boundary check
if (idx >= n) return;

// in-place reduction in global memory
for (int stride = 1; stride < blockDim.x; stride *= 2)
{
if ((tid % (2 * stride)) == 0)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}