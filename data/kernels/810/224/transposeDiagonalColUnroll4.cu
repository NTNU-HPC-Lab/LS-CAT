#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalColUnroll4(float *out, float *in, const int nx, const int ny)
{
unsigned int blk_y = blockIdx.x;
unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

unsigned int ix_stride = blockDim.x * blk_x;
unsigned int ix = ix_stride * 4 + threadIdx.x;
unsigned int iy = blockDim.y * blk_y + threadIdx.y;

if (ix < nx && iy < ny)
{
out[iy * nx + ix] = in[ix * ny + iy];
out[iy * nx + ix + blockDim.x] = in[(ix + blockDim.x) * ny + iy];
out[iy * nx + ix + 2 * blockDim.x] =
in[(ix + 2 * blockDim.x) * ny + iy];
out[iy * nx + ix + 3 * blockDim.x] =
in[(ix + 3 * blockDim.x) * ny + iy];
}
}