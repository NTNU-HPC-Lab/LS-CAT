#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
// global to slice : global grid idx for (x,y) plane
unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

// smem idx for current point
unsigned int stx = threadIdx.x + NPAD;
unsigned int idx  = ix + iStart * nx;

// shared memory for u2 with size [4+16+4][4+16+4]
__shared__ float tile[BDIMX + NPAD2];

const float alpha = 0.12f;

// register for y value
float yval[9];

for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

// to be used in z loop
int iskip = NPAD * nx;

#pragma unroll 9
for (int iy = iStart; iy < iEnd; iy++)
{
// get front3 here
yval[8] = g_u2[idx + iskip];

if(threadIdx.x < NPAD)
{
tile[threadIdx.x]  = g_u2[idx - NPAD];
tile[stx + BDIMX]    = g_u2[idx + BDIMX];
}

tile[stx] = yval[4];
__syncthreads();

if ( (ix >= NPAD) && (ix < nx - NPAD) )
{
// 8rd fd operator
float tmp = coef[0] * tile[stx] * 2.0f;

#pragma unroll
for(int d = 1; d <= 4; d++)
{
tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
}

#pragma unroll
for(int d = 1; d <= 4; d++)
{
tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
}

// time dimension
g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
}

#pragma unroll 8
for (int i = 0; i < 8 ; i++)
{
yval[i] = yval[i + 1];
}

// advancd on global idx
idx  += nx;
__syncthreads();
}
}