#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = i + offset;

if (k < n) C[k] = A[i] + B[i];
}