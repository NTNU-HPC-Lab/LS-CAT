#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid >= N) return;

int old = *shared_var;
*shared_var = old + 1;
values_read[tid] = old;

for (i = 0; i < iters; i++)
{
int old = *shared_var;
*shared_var = old + 1;
}
}