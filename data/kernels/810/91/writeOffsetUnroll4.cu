#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = i + offset;

if (k + 3 * blockDim.x < n)
{
C[k]              = A[i]              + B[i];
C[k + blockDim.x]   = A[i +  blockDim.x] + B[i +  blockDim.x];
C[k + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x];
C[k + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x];
}
}