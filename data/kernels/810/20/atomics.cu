#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid >= N) return;

values_read[tid] = atomicAdd(shared_var, 1);

for (i = 0; i < iters; i++)
{
atomicAdd(shared_var, 1);
}
}