#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemUnrollPad(float *out, float *in, const int nx, const int ny)
{
// static 1D shared memory with padding
__shared__ float tile[BDIMY * (BDIMX * 2 + IPAD)];

// coordinate in original matrix
unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

// linear global memory index for original matrix
unsigned int ti = iy * nx + ix;

// thread index in transposed block
unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
unsigned int irow = bidx / blockDim.y;
unsigned int icol = bidx % blockDim.y;

// coordinate in transposed matrix
unsigned int ix2 = blockIdx.y * blockDim.y + icol;
unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

// linear global memory index for transposed matrix
unsigned int to = iy2 * ny + ix2;

if (ix + blockDim.x < nx && iy < ny)
{
// load two rows from global memory to shared memory
unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) +
threadIdx.x;
tile[row_idx]         = in[ti];
tile[row_idx + BDIMX] = in[ti + BDIMX];

// thread synchronization
__syncthreads();

// store two rows to global memory from two columns of shared memory
unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
out[to] = tile[col_idx];
out[to + ny * BDIMX] = tile[col_idx + BDIMX];
}
}