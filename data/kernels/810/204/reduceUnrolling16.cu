#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling16 (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 16 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 16;

// unrolling 16
if (idx + 15 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
int b1 = g_idata[idx + 4 * blockDim.x];
int b2 = g_idata[idx + 5 * blockDim.x];
int b3 = g_idata[idx + 6 * blockDim.x];
int b4 = g_idata[idx + 7 * blockDim.x];
int c1 = g_idata[idx + 8 * blockDim.x];
int c2 = g_idata[idx + 9 * blockDim.x];
int c3 = g_idata[idx + 10 * blockDim.x];
int c4 = g_idata[idx + 11 * blockDim.x];
int d1 = g_idata[idx + 12 * blockDim.x];
int d2 = g_idata[idx + 13 * blockDim.x];
int d3 = g_idata[idx + 14 * blockDim.x];
int d4 = g_idata[idx + 15 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4 + c1 + c2 + c3 + c4
+ d1 + d2 + d3 + d4;
}

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}