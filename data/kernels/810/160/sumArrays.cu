#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < N)
{
for (int i = 0; i < N; ++i)
{
C[idx] = A[idx] + B[idx];
}
}
}