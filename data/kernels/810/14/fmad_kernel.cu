#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fmad_kernel(double x, double y, double *out)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if (tid == 0)
{
*out = x * x + y;
}
}