#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = i + offset;

if (k < n) C[k] = A[k] + B[k];
}