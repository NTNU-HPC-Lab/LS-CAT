#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps (int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 2;

// unrolling 2
if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];

__syncthreads();

// in-place reduction in global memory
for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
{
if (tid < stride)
{
idata[tid] += idata[tid + stride];
}

// synchronize within threadblock
__syncthreads();
}

// unrolling last warp
if (tid < 32)
{
volatile int *vsmem = idata;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

if (tid == 0) g_odata[blockIdx.x] = idata[0];
}