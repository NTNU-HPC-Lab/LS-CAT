#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
extern __shared__ int smem[];

// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

// unrolling 4
int tmpSum = 0;

if (idx + 3 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
tmpSum = a1 + a2 + a3 + a4;
}

smem[tid] = tmpSum;
__syncthreads();

// in-place reduction in global memory
if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vsmem = smem;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = smem[0];
}