#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readWriteOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
unsigned int k = i + offset;

if (k + 3 * blockDim.x < n)
{
C[k]                  = A[k]                  + B[k];
C[k + blockDim.x]     = A[k + blockDim.x]     + B[k + blockDim.x];
C[k + 2 * blockDim.x] = A[k + 2 * blockDim.x] + B[k + 2 * blockDim.x];
C[k + 3 * blockDim.x] = A[k + 3 * blockDim.x] + B[k + 3 * blockDim.x];
}
}