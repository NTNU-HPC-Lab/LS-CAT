#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int iy = blockIdx.y;
unsigned int idx = iy * nx + ix;

if (ix < nx && iy < ny)
MatC[idx] = MatA[idx] + MatB[idx];
}