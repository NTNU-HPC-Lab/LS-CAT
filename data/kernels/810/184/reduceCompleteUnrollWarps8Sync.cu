#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata, unsigned int n)
{
// set thread ID
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x * 8;

// unrolling 8
if (idx + 7 * blockDim.x < n)
{
int a1 = g_idata[idx];
int a2 = g_idata[idx + blockDim.x];
int a3 = g_idata[idx + 2 * blockDim.x];
int a4 = g_idata[idx + 3 * blockDim.x];
int b1 = g_idata[idx + 4 * blockDim.x];
int b2 = g_idata[idx + 5 * blockDim.x];
int b3 = g_idata[idx + 6 * blockDim.x];
int b4 = g_idata[idx + 7 * blockDim.x];
g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
}

__syncthreads();

// in-place reduction and complete unroll
if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
idata[tid] += idata[tid + 32];
__syncthreads();

if (tid < 32)
idata[tid] += idata[tid + 16];
__syncthreads();

if (tid < 32)
idata[tid] += idata[tid +  8];
__syncthreads();

if (tid < 32)
idata[tid] += idata[tid +  4];
__syncthreads();

if (tid < 32)
idata[tid] += idata[tid +  2];
__syncthreads();

if (tid < 32)
idata[tid] += idata[tid +  1];

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = idata[0];
}