#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
__shared__ int smem[DIM];

// set thread ID
unsigned int tid = threadIdx.x;

// boundary check
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx >= n) return;

// convert global data pointer to the local pointer of this block
int *idata = g_idata + blockIdx.x * blockDim.x;

// set to smem by each threads
smem[tid] = idata[tid];
__syncthreads();

// in-place reduction in shared memory
if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

__syncthreads();

if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

__syncthreads();

if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

__syncthreads();

if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];

__syncthreads();

// unrolling warp
if (tid < 32)
{
volatile int *vsmem = smem;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid +  8];
vsmem[tid] += vsmem[tid +  4];
vsmem[tid] += vsmem[tid +  2];
vsmem[tid] += vsmem[tid +  1];
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = smem[0];
}