#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *g_data, float value)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
g_data[idx] = g_data[idx] + value;
}