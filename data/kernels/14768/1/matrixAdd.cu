#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd(int *a,int *b,int *c)
{
int col=blockIdx.x*blockDim.x+threadIdx.x;
int row=blockIdx.y*blockDim.y+threadIdx.y;
int index=col+row*N;
printf("\n%d\t%d",threadIdx.x,threadIdx.y);
printf("\nIndex val:%d\n",index);
if(col<N && row<N)
{
c[index]=a[index]+b[index];
}
}