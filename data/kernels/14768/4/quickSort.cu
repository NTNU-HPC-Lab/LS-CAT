#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void quickSort(int *x, int *dfirst, int *dlast, int *list)
{
int idx = threadIdx.x;
int first = dfirst[idx];
int last = dlast[idx];
list[idx] = 0;

if(first<last)
{
int pivot, j, temp, i;

pivot = first;
i = first;
j = last;

while(i<j)
{
while(x[i]<=x[pivot] && i<last)
i++;
while(x[j] > x[pivot])
j--;
if(i<j)
{
temp = x[i];
x[i] = x[j];
x[j] = temp;
}
}

temp = x[pivot];
x[pivot] = x[j];
x[j] = temp;

for(i=first; i<=last; i++)
if(x[i] > x[i+1])
{
list[idx] = j+1;
break;
}
}
}