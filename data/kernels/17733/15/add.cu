#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
int tid = blockIdx.x;    // this thread handles the data at its thread id
if (tid < N)
c[tid] = a[tid] + b[tid];
}