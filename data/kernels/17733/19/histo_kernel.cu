#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {
// calculate the starting index and the offset to the next
// block that each thread will be processing
int i = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while (i < size) {
atomicAdd( &histo[buffer[i]], 1 );
i += stride;
}
}