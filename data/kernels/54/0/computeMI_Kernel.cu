#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t calcCuda(float *output, const uint8_t *input, const size_t *size);

//********************************************************************************************\\
static hipDeviceProp_t deviceProperties_;
__global__ void computeMI_Kernel(float *MIs, uint8_t *input, int rowCount, int colCount, int *countNZ, int offset)
{
int i = threadIdx.x + blockIdx.x * blockDim.x + offset;
if (i > rowCount*(rowCount-1)/2) return;
int joints[2][2] = { 0 };
int countNZA , countNZB, a, b, j, k;
float joint;
uint8_t *inputA = 0,
*inputB = 0;

MIs += i;
a = 1;
b = 0;
for (j = rowCount - 1; j > 1; j--)
{
if (i < j) break;

a++;
b++;
i -= j;
}
j = b;
i += a;

//  MIs += i + j*rowCount;
*MIs = 0;
// *MIs = i * 1000 + j;

//  for (j = 0; j < i; j++, MIs += colCount)
{

inputA = input + i;
inputB = input + j;
countNZA = countNZ[i];
countNZB = countNZ[j];
for (k = 0; k < colCount; k++, inputA += rowCount, inputB += rowCount)
{
joints[*inputA][*inputB]++;
}

for (k = 0; k < 4; k++)
{
a = k % 2;
b = k / 2;

joint = joints[a][b];
if (joint == 0)
continue;
joint /= colCount;
if (a) a = countNZA;
else a = colCount - countNZA;
if (b) b = countNZB;
else b = colCount - countNZB;

*MIs += joint * log2f(joint / ((float)a / colCount) / ((float)b / colCount));
}
}
/*  size_t i, t_count, b_count;
t_count = rowCount > deviceProperties_.maxThreadsPerBlock ? deviceProperties_.maxThreadsPerBlock : rowCount;
b_count = rowCount / deviceProperties_.maxThreadsPerBlock + 1;
computeMI_Kernel << <b_count, t_count >> > (MIs, input, i, rowCount, colCount, countNZ)
*/
}//********************************************************************************************\\