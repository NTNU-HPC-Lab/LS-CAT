#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "gLSTMOutputBackward.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *outCell = NULL;
hipMalloc(&outCell, XSIZE*YSIZE);
float *outXW = NULL;
hipMalloc(&outXW, XSIZE*YSIZE);
float *outSU = NULL;
hipMalloc(&outSU, XSIZE*YSIZE);
float *outB = NULL;
hipMalloc(&outB, XSIZE*YSIZE);
const float *cell = NULL;
hipMalloc(&cell, XSIZE*YSIZE);
const float *xW = NULL;
hipMalloc(&xW, XSIZE*YSIZE);
const float *sU = NULL;
hipMalloc(&sU, XSIZE*YSIZE);
const float *b = NULL;
hipMalloc(&b, XSIZE*YSIZE);
const float *adj = NULL;
hipMalloc(&adj, XSIZE*YSIZE);
size_t rows = XSIZE;
size_t cols = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gLSTMOutputBackward<<<gridBlock,threadBlock>>>(outCell,outXW,outSU,outB,cell,xW,sU,b,adj,rows,cols);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
gLSTMOutputBackward<<<gridBlock,threadBlock>>>(outCell,outXW,outSU,outB,cell,xW,sU,b,adj,rows,cols);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
gLSTMOutputBackward<<<gridBlock,threadBlock>>>(outCell,outXW,outSU,outB,cell,xW,sU,b,adj,rows,cols);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}