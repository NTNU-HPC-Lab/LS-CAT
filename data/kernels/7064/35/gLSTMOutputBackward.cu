#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableLogit(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gLSTMOutputBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* adj, size_t rows, size_t cols) {
for(int bid = 0; bid < rows; bid += gridDim.x) {
int j = bid + blockIdx.x;
if(j < rows) {
float* rowOutCell = outCell + j * cols;
float* rowOutXW = outXW + j * cols * 4;
float* rowOutSU = outSU + j * cols * 4;

const float* rowCell = cell + j * cols;
const float* xWrow = xW + j * cols * 4;
const float* sUrow = sU + j * cols * 4;

const float* rowAdj = adj + j * cols;

for(int tid = 0; tid < cols; tid += blockDim.x) {
int i = tid + threadIdx.x;
if(i < cols) {
int k = i + 3 * cols;
float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);

float t = tanhf(rowCell[i]);

float adj = rowAdj[i];

// dc/dc_{t-1}
if(outCell)
rowOutCell[i] += go * (1 - t * t) * adj;

// dc/d(b_o) = dc/d(xW_f) ...
float dcdxo = t * go * (1 - go) * adj;
if(outXW)
rowOutXW[k] += dcdxo;
if(outSU)
rowOutSU[k] += dcdxo;
if(outB)
atomicAdd(outB + k, dcdxo);
}
}
}
}
}