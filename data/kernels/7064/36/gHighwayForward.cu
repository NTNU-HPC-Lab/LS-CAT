#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableLogit(float x) {
if(x >= 0) {
float z = expf(-x);
return 1.0 / (1.0 + z);
} else {
float z = expf(x);
return z / (1.0 + z);
}
}
__global__ void gHighwayForward(float* out, const float* in1, const float* in2, const float* t, size_t length) {
for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
if(index < length) {
float sigma = stableLogit(t[index]);
out[index] = in1[index] * sigma + in2[index] * (1.f - sigma);
}
}
}