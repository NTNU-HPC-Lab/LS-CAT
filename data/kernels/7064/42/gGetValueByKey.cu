#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
int tid = threadIdx.x + blockDim.x * blockIdx.x;
if(tid < n) {
int index = indeces[tid];
d_out[tid] = d_in[index];
}
}