#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 32


__global__ void derivativeError(float *output, float *actual, float *deriv_err)
{
__shared__ float sdata[1024];

//ideally block is 1024x1 and grid is ??? x units
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y;

sdata[threadIdx.x] = output[row*gridDim.y + col];
__syncthreads();

for(int s= blockDim.x / 2; s>0; s>>=1)
{
if(threadIdx.x < s)
sdata[threadIdx.x] += sdata[threadIdx.x+s];
__syncthreads();
}
if(threadIdx.x == 0) //only tid0 can write
{
/*deriv_err[blockIdx.x] = sdata[0]*/deriv_err[blockDim.y*blockIdx.x+col] = sdata[blockIdx.x];
}
}