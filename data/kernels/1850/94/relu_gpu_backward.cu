#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < N)
ingrad[tid] = indata[tid] > 0 ? 1 * outgrad[tid] : 0;
}