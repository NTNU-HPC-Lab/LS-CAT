#include "hip/hip_runtime.h"
#include "includes.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render(float *fb, int max_x, int max_y) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;
if((i >= max_x) || (j >= max_y)) return;
int pixel_index = j*max_x*3 + i*3;
fb[pixel_index + 0] = float(i) / max_x;
fb[pixel_index + 1] = float(j) / max_y;
fb[pixel_index + 2] = 0.0;
}