#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_histogram(char* dbuff, unsigned int* dcount, unsigned int size, float stride) {

unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int start_pos = stride * index;
unsigned int stop_pos = start_pos + stride;
unsigned int lcount[10] = { 0 };

if (size < stop_pos) {
stop_pos = size;
}

for (unsigned int i = start_pos; i < stop_pos; i++) {
// Increment counter per occurances
if (dbuff[i] == '0') {
lcount[0] += 1;
} else if (dbuff[i] == '1') {
lcount[1] += 1;
} else if (dbuff[i] == '2') {
lcount[2] += 1;
} else if (dbuff[i] == '3') {
lcount[3] += 1;
} else if (dbuff[i] == '4') {
lcount[4] += 1;
} else if (dbuff[i] == '5') {
lcount[5] += 1;
} else if (dbuff[i] == '6') {
lcount[6] += 1;
} else if (dbuff[i] == '7') {
lcount[7] += 1;
} else if (dbuff[i] == '8') {
lcount[8] += 1;
} else if (dbuff[i] == '9') {
lcount[9] += 1;
}
}

__syncthreads();

dcount[0] += lcount[0];
dcount[1] += lcount[1];
dcount[2] += lcount[2];
dcount[3] += lcount[3];
dcount[4] += lcount[4];
dcount[5] += lcount[5];
dcount[6] += lcount[6];
dcount[7] += lcount[7];
dcount[8] += lcount[8];
dcount[9] += lcount[9];
}