#include "hip/hip_runtime.h"
#include "includes.h"

static const int n_el = 512;
static const size_t size = n_el * sizeof(float);
// declare the kernel function


// function which invokes the kernel
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
// calculate the unique thread index
int tid = blockDim.x * blockIdx.x + threadIdx.x;
// perform tid-th elements addition
if (tid < n_el) C[tid] = A[tid] + B[tid];
}