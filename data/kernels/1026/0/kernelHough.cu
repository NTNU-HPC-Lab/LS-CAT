#include "hip/hip_runtime.h"
#include "includes.h"

#define iceil(num, den) (num + den - 1) / den
#define ARRAY_SIZE 20 //must be an even number; this number/2 = number of points //sets random array and constant mem size
//#define BIN 100 //divides the grid into square bins to vote on. perfect square value
#define NUM_LINES 4 //top X voted lines. Picks first X Largest from top left to bottom right of grid space.

/*GRID evaluated for bin voting
* Must always be a square grid with origin at center
*/
#define dimension 5
#define LXBOUND (-1*dimension) //lowest X
#define RXBOUND (dimension) //highest X
#define LYBOUND (-1*dimension) //lowest Y
#define UYBOUND (dimension) //highest Y
////////////////////////////////

#define INCREMENT 1 //precision, length of 1 side of the square(bin)
//The (abs)difference between between two sides is the length of the grid. Length/Increment determines how many bins

#define column (((RXBOUND - LXBOUND) / INCREMENT) * ((RXBOUND - LXBOUND) / INCREMENT)) / ((RXBOUND + UYBOUND) / INCREMENT)

__constant__ int d_coordarray[ARRAY_SIZE];//Place coordinates in constant memory

//show grid with votes. Becomes unuseful when bins > 20x20
__global__ void kernelHough(int size, int* d_binarray) {
/*
take a piece of the array. discretize into y=mx+b format per point. check all points and increment all bins touched
at the end recombine all shared memory to a global bin tally. Take the most significant X numbers as lines.
discretized from point(1,1) ==(m,n)==> (-1,1)
check each bin for count and sum them to a global array in sync
NUM of coordinates will check all bins for their own equation and increment appropriately
*/

// Number from 0 through arraysize / 2
const int thread = 2 * (blockDim.x * blockIdx.x + threadIdx.x);

// Slope is discretized space = -x
const float slope = -1.0 * d_coordarray[thread];

// Intercept in discretized space = y
const float intercept = d_coordarray[thread + 1];

int counter = 0;//keeps current array index being checked
//loop through entire graph
for (float x = LXBOUND; x < RXBOUND; x += INCREMENT) {

const float xMin = x;
const float xMax = x + INCREMENT;

for (float y = UYBOUND; y > LYBOUND; y -= INCREMENT) {

const float yMin = y - INCREMENT;
const float yMax = y;
//calculates possible y range associated with the known x range
const float lower_range = slope * xMin + intercept;
const float upper_range = slope * xMax + intercept;
//if the possible y ranges corresponding to the x values exist within the actual y range increment bin
if ((lower_range <= yMax && lower_range >= yMin) || (upper_range <= yMax && upper_range >= yMin))
atomicAdd(&d_binarray[counter], 1);//increment bin, protected from race condition

counter++;
}
}
}