#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{   long int initIteration, endIteration;
long int i = 0;
double piPartial;

int index = (blockDim.x * blockIdx.x) + threadIdx.x;

initIteration = (iterations/totalThreads) * index;
endIteration = initIteration + (iterations/totalThreads) - 1;

i = initIteration;
piPartial = 0;

do{
piPartial = piPartial + (double)(4.0 / ((i*2)+1));
i++;
piPartial = piPartial - (double)(4.0 / ((i*2)+1));
i++;
}while(i < endIteration);

piTotal[index] = piPartial;

__syncthreads();
if(index == 0){
for(i = 1; i < totalThreads; i++)
piTotal[0] = piTotal[0] + piTotal[i];
}
}