#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copySharedMem(float *odata, float *idata, int width, int height)
{
__shared__ float tile[TILE_DIM][TILE_DIM];

int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

int index  = xIndex + width*yIndex;

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
if (xIndex < width && yIndex < height)
{
tile[threadIdx.y][threadIdx.x] = idata[index];
}
}

__syncthreads();

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
if (xIndex < height && yIndex < width)
{
odata[index] = tile[threadIdx.y][threadIdx.x];
}
}
}