#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeFineGrained(float *odata, float *idata, int width, int height)
{
__shared__ float block[TILE_DIM][TILE_DIM+1];

int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
int index = xIndex + (yIndex)*width;

for (int i=0; i < TILE_DIM; i += BLOCK_ROWS)
{
block[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
}

__syncthreads();

for (int i=0; i < TILE_DIM; i += BLOCK_ROWS)
{
odata[index+i*height] = block[threadIdx.x][threadIdx.y+i];
}
}