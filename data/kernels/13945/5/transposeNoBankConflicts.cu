#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height)
{
__shared__ float tile[TILE_DIM][TILE_DIM+1];

int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
int index_in = xIndex + (yIndex)*width;

xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
int index_out = xIndex + (yIndex)*height;

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
}

__syncthreads();

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
}
}