#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonal(float *odata, float *idata, int width, int height)
{
__shared__ float tile[TILE_DIM][TILE_DIM+1];

int blockIdx_x, blockIdx_y;

// do diagonal reordering
if (width == height)
{
blockIdx_y = blockIdx.x;
blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
}
else
{
int bid = blockIdx.x + gridDim.x*blockIdx.y;
blockIdx_y = bid%gridDim.y;
blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
}

// from here on the code is same as previous kernel except blockIdx_x replaces blockIdx.x
// and similarly for y

int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;
int index_in = xIndex + (yIndex)*width;

xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
int index_out = xIndex + (yIndex)*height;

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
}

__syncthreads();

for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
{
odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
}
}