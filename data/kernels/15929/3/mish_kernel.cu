#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float softplus_kernel(float x, float threshold = 20) {
if (x > threshold) return x;                // too large
else if (x < -threshold) return expf(x);    // too small
return logf(expf(x) + 1);
}
__device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}
__global__ void mish_kernel(const float *input, float *output, int num_elem) {

int idx = threadIdx.x + blockDim.x * blockIdx.x;
if (idx >= num_elem) return;

//float t = exp(input[idx]);
//if (input[idx] > 20.0) {
//    t *= t;
//    output[idx] = (t - 1.0) / (t + 1.0);
//} else {
//    float tt = t * t;
//    output[idx] = (tt + 2.0 * t) / (tt + 2.0 * t + 2.0);
//}
//output[idx] *= input[idx];
output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
}