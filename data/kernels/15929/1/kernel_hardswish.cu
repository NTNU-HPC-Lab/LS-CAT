#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hardswish(const float *input_, float *output_, int n_data_size_)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i >= n_data_size_)return;
if (input_[i] >= 3.0f)
{
output_[i] = input_[i];
}
else if (input_[i] <= -3.0f)
{
output_[i] = 0.0f;
}
else
{
output_[i] = input_[i] * (input_[i] + 3.0f) / 6.0f;
}
}