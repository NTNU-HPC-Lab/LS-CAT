#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void add(std::size_t  n, const float *x, float *y) {
std::size_t  index = blockIdx.x * blockDim.x + threadIdx.x;
std::size_t  stride = blockDim.x * gridDim.x;
for (auto i = index; i < n; i += stride) y[i] = x[i] + y[i];
}