#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Thumbnail_uchar(hipTextureObject_t uchar_tex, int *histogram, int src_width, int src_height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
if (y < src_height && x < src_width)
{
unsigned char pixel = tex2D<unsigned char>(uchar_tex, x, y);
atomicAdd(&histogram[pixel], 1);
}
}