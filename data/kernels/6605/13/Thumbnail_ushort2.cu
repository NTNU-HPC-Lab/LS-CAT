#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Thumbnail_ushort2(hipTextureObject_t ushort2_tex, int *histogram, int src_width, int src_height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y < src_height && x < src_width)
{
ushort2 pixel = tex2D<ushort2>(ushort2_tex, x, y);
atomicAdd(&histogram[(pixel.x + 128) >> 8], 1);
atomicAdd(&histogram[256 + (pixel.y + 128) >> 8], 1);
}
}