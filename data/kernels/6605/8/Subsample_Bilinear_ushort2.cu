#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_ushort2(hipTextureObject_t ushort2_tex, ushort2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
ushort2 c0 = tex2D<ushort2>(ushort2_tex, xi-dx, yi-dy);
ushort2 c1 = tex2D<ushort2>(ushort2_tex, xi+dx, yi-dy);
ushort2 c2 = tex2D<ushort2>(ushort2_tex, xi-dx, yi+dy);
ushort2 c3 = tex2D<ushort2>(ushort2_tex, xi+dx, yi+dy);
int2 uv;
uv.x = ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
uv.y = ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
dst[yo*dst_pitch2+xo] = make_ushort2((unsigned short)uv.x, (unsigned short)uv.y);
}
}