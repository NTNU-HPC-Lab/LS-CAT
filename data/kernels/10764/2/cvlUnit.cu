#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core, char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int index=y*lenX+x;
if(x>=lenX||y>=lenY)return;
if(x-lenCore/2<0||x+lenCore/2>=lenX||y-lenCore/2<0||y+lenCore/2>=lenY){
outR[index]=imgR[index];
outG[index]=imgG[index];
outB[index]=imgB[index];
return ;
}


int i,j,tmpX,tmpY;
int sumR=0;
int sumG=0;
int sumB=0;
for(i=0;i<lenCore;i++){
for(j=0;j<lenCore;j++){
tmpX = x-lenCore/2+i;
tmpY = y-lenCore/2+j;
//			if(x==8&&y==8){printf("tmpX=%d,tmpY=%d:\n",tmpX,tmpY);}
sumR+=imgR[tmpY*lenX+tmpX]*core[j*lenCore+i];
//			if(x==8&&y==8){
//				printf("\tR:\t %d*%d,new=%d\n",imgR[tmpY*lenX+tmpX],core[j*lenCore+i],sumR);
//			}
sumG+=imgG[tmpY*lenX+tmpX]*core[j*lenCore+i];
//			if(x==8&&y==8){
//				printf("\tG:\t %d*%d,new=%d\n",imgG[tmpY*lenX+tmpX],core[j*lenCore+i],sumG);
//			}
sumB+=imgB[tmpY*lenX+tmpX]*core[j*lenCore+i];
//			if(x==8&&y==8){
//				printf("\tB:\t %d*%d,new=%d\n",imgB[tmpY*lenX+tmpX],core[j*lenCore+i],sumB);
//			}
}
}
outR[index]=(char)(sumR*1.0/(lenCore*lenCore));
outG[index]=(char)(sumG*1.0/(lenCore*lenCore));
outB[index]=(char)(sumB*1.0/(lenCore*lenCore));
return;
}