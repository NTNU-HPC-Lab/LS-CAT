#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TopForcing(double ppt, double *eff_rain, int size) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
while (tid < size) {
eff_rain[tid] = ppt;
tid += blockDim.x * gridDim.x;
}
}