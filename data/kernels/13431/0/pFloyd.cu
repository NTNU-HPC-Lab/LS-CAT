#include "hip/hip_runtime.h"
#include "includes.h"
/****************************************************************************
Floyd - Warshall Algorithm developed using CUDA. A 2011-2012 assignement for
Parallel Programming Course of Electrical and Computer Engineering Department
in the Aristotle Faculty of Enginnering - Thessaloniki.

*****************************************************************************/


#define INF 100000000
#define BLOCKSIZE 128
#define BITSFT 7 //log2(BLOCKSIZE)


/*****************************************
Array Generator - filling weight matrices
according to Floyd-Warshall theory.
******************************************/
__global__ void pFloyd(float *D,float *Q,int vertices,int k,int k2)
{
int i,j,index;
i= blockIdx.x;
j=(blockIdx.y << BITSFT) + threadIdx.x;
index=(i << vertices)+j; 				//vertices equals log2(vertices).
if((D[(i << vertices)+k]+D[(k2)+j])<D[index])
{
D[index]=D[(i << vertices)+k]+D[(k2)+j];
Q[index]=k;
}
}