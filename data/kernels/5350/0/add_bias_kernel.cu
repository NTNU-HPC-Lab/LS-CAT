#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_KERNEL_LOOP(i ,n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i<(n); i+= blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

__global__ void add_bias_kernel( int n, float* data_out, const float* bias, const int out_channels, const int height_out, const int width_out ){
CUDA_KERNEL_LOOP(index, n){
const int c_col = (index / width_out / height_out) % out_channels;
float value = bias[c_col];
atomicAdd(data_out + index, value);
}
}