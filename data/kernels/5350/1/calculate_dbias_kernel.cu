#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_KERNEL_LOOP(i ,n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i<(n); i+= blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

__global__ void calculate_dbias_kernel( int n, const float* grad_output, float* grad_bias, const int out_channels, const int height_out, const int width_out ){
CUDA_KERNEL_LOOP(index, n){
const int c_col = (index / width_out / height_out) % out_channels;
float value = grad_output[index];
atomicAdd(grad_bias + c_col, value);
}
}