#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels




__global__
__global__ void vectorScaling(const double *A, double s, double *C, int numElements)
{
int gridIndex = blockDim.x * blockIdx.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;

for (int i = gridIndex; i<numElements; i+=stride)
{
C[i] = A[i]*s;
}
}