#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void total(float *input, float *output, int len){
__shared__ float partialSum[2*BLOCK_SIZE];
unsigned int t=threadIdx.x,start=2*blockIdx.x*BLOCK_SIZE;

if(start+t<len)	partialSum[t] = input[start+t];

else partialSum[t]=0;
__syncthreads();
if(start+BLOCK_SIZE+t<len)partialSum[BLOCK_SIZE+t]=input[start+BLOCK_SIZE+t];
else partialSum[BLOCK_SIZE+t]=0;
__syncthreads();
for(unsigned int stride=BLOCK_SIZE;stride>=1; stride>>=1){
__syncthreads();
if (t<stride) partialSum[t]+=partialSum[t+stride];
__syncthreads();
}
if(t==0) output[blockIdx.x]=partialSum[0];
}