#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels




__global__
__global__ void matrixTransposeSqr(double *P, double* M, int width, int height)
{
unsigned int xIdx = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int yIdx = blockDim.y * blockIdx.y + threadIdx.y;

if (xIdx < width && yIdx < height)
{
unsigned int inIdx  = xIdx + width * yIdx;
unsigned int outIdx= yIdx + height * xIdx;
P[outIdx] = M[inIdx];
}
}