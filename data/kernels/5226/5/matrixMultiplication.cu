#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels




__global__
__global__ void matrixMultiplication(double* M, double* N, double* P, int widthAHeightB, int heightA, int widthB) {

int Mstart=widthAHeightB*tileSize*blockIdx.y;
int Mend=Mstart+ widthAHeightB - 1;
int mstep=tileSize;
int Nstart=tileSize*blockIdx.x;
int nstep=tileSize*widthB;
double temp=0;

__shared__ double Ms[tileSize][tileSize];
__shared__ double Ns[tileSize][tileSize];

//area where the tiles fits without "cutting"
if(Mstart < (heightA/tileSize)*tileSize*widthAHeightB && Nstart%widthB < (widthB/tileSize)*tileSize ){
for(int m=Mstart,n=Nstart;m<Mend;m+=mstep,n+=nstep){
Ms[threadIdx.y][threadIdx.x]=M[m+widthAHeightB*threadIdx.y+threadIdx.x];
Ns[threadIdx.y][threadIdx.x]=N[n+widthB*threadIdx.y+threadIdx.x];
__syncthreads();


for (int i = 0; i < tileSize; ++i) {
temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
}
__syncthreads();

}
} else {//the rest of the matrix
for(int m=Mstart,n=Nstart;m<=Mend;m+=mstep,n+=nstep){

if(m%widthAHeightB + threadIdx.x < widthAHeightB && blockIdx.y*tileSize + threadIdx.y < heightA){
Ms[threadIdx.y][threadIdx.x]=M[m+widthAHeightB*threadIdx.y+threadIdx.x];
}
else{
Ms[threadIdx.y][threadIdx.x]=0.0;
}

if((n/widthB) + threadIdx.y < widthAHeightB && blockIdx.x*tileSize + threadIdx.x < widthB){
Ns[threadIdx.y][threadIdx.x]=N[n+widthB*threadIdx.y+threadIdx.x];
}
else{
Ns[threadIdx.y][threadIdx.x]=0.0;
}
__syncthreads();


for (int i = 0; i < tileSize; ++i) {
temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
}
__syncthreads();

}
}



if(blockIdx.y*tileSize + threadIdx.y < heightA && blockIdx.x*tileSize + threadIdx.x < widthB){
P[widthB * tileSize * blockIdx.y + tileSize * blockIdx.x + widthB * threadIdx.y + threadIdx.x] = temp;
}
}