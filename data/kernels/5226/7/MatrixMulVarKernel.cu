#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulVarKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {
int Row = blockIdx.y*blockDim.y+threadIdx.y;// Calculate the row index of the P element and M
int Col = blockIdx.x*blockDim.x+threadIdx.x;// Calculate the column index of P and N
if ((Row < heightA) && (Col < widthB)) {
float Pvalue = 0;
for (int k = 0; k < widthAHeightB; ++k) {
Pvalue += M[Row*widthAHeightB+k]*N[k*widthB+Col];// each thread computes one element of the block sub-matrix
}

P[Row*widthB+Col] = Pvalue;
}
}