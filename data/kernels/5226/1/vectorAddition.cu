#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels




__global__
__global__ void vectorAddition(const double *A, const double *B, double *C, int numElements)
{
int gridIndex = blockDim.x * blockIdx.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;

for (int i = gridIndex; i<numElements; i+=stride)
{
C[i] = A[i] + B[i];
}
}