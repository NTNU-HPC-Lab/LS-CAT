#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulVarSharedMemoryKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {

int Mstart=widthAHeightB*tileSize*blockIdx.y;
int Mend=Mstart+ widthAHeightB - 1;
int mstep=tileSize;
int Nstart=tileSize*blockIdx.x;
int nstep=tileSize*widthB;
float temp=0;

__shared__ float Ms[tileSize][tileSize];
__shared__ float Ns[tileSize][tileSize];

//area where the tiles fits without "cutting"
if(Mstart < (heightA/tileSize)*tileSize*widthAHeightB && Nstart%widthB < (widthB/tileSize)*tileSize ){
for(int m=Mstart,n=Nstart;m<Mend;m+=mstep,n+=nstep){
Ms[threadIdx.y][threadIdx.x]=M[m+widthAHeightB*threadIdx.y+threadIdx.x];
Ns[threadIdx.y][threadIdx.x]=N[n+widthB*threadIdx.y+threadIdx.x];
__syncthreads();


for (int i = 0; i < tileSize; ++i) {
temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
}
__syncthreads();

}
} else {//the rest of the matrix
for(int m=Mstart,n=Nstart;m<=Mend;m+=mstep,n+=nstep){

if(m%widthAHeightB + threadIdx.x < widthAHeightB && blockIdx.y*tileSize + threadIdx.y < heightA){
Ms[threadIdx.y][threadIdx.x]=M[m+widthAHeightB*threadIdx.y+threadIdx.x];
}
else{
Ms[threadIdx.y][threadIdx.x]=0.0;
}

if((n/widthB) + threadIdx.y < widthAHeightB && blockIdx.x*tileSize + threadIdx.x < widthB){
Ns[threadIdx.y][threadIdx.x]=N[n+widthB*threadIdx.y+threadIdx.x];
}
else{
Ns[threadIdx.y][threadIdx.x]=0.0;
}
__syncthreads();


for (int i = 0; i < tileSize; ++i) {
temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
}
__syncthreads();

}
}



if(blockIdx.y*tileSize + threadIdx.y < heightA && blockIdx.x*tileSize + threadIdx.x < widthB){
P[widthB * tileSize * blockIdx.y + tileSize * blockIdx.x + widthB * threadIdx.y + threadIdx.x] = temp;
}
}