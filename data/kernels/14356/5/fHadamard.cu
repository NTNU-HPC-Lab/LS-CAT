#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"
__global__ void fHadamard( const float* argumentsA, const float* argumentsB, float* results, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
results[index] = argumentsA[index] * argumentsB[index];
}
}