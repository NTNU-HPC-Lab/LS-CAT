#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void dropoutTest( const float* arguments, float* results, const float dropoutFraction, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
results[index] = arguments[index] * (1.f - dropoutFraction);
}
}