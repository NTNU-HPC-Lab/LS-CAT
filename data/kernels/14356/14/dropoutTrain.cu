#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void dropoutTrain( const float* arguments, float* dropoutMask, float* results, const float dropoutFraction, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
const float mask = dropoutFraction < dropoutMask[index];
dropoutMask[index] = mask;
results[index] = mask * arguments[index];
}
}