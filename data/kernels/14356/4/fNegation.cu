#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"
__global__ void fNegation( const float* arguments, float* results, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
results[index] = -arguments[index];
}
}