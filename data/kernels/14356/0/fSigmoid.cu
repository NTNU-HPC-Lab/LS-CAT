#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"
__global__ void fSigmoid( const float* arguments, float* results, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
results[index] = 1.f / (1.f + expf(-arguments[index]));
}
}