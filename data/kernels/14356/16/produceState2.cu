#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
return 1 / (1 + expf(-x));
}
__global__ void produceState2(const float* arguments, const int argsSize, const float* weights, const int* topology, const int topSize, float* outStates) {
const int tid = threadIdx.x;
const int dim = argsSize + topSize;
extern __shared__ float s[];
float* states = s;
bool* ready = (bool*)&states[dim];
__shared__ int counter[1];

int r = tid;
while(r < dim) {
ready[r] = false;
r += blockDim.x;
}


if (tid == 0) {
counter[tid] = argsSize;
}
if (tid < argsSize) {
states[tid] = arguments[tid];
ready[tid] = true;
}
__syncthreads();

while(counter[0] < dim) {
const int index = counter[0] + tid;
const int topIndex = index - argsSize;
if (topIndex < topSize) {
const int leftBorder = topology[topIndex*3];
const int rightBorder = topology[topIndex*3 + 1];
const int weightsStart = topology[topIndex*3 + 2];

if (rightBorder <= counter[0]) {
float sum = 0;
for (int i = leftBorder; i < rightBorder; i++) {
sum += states[i] * weights[weightsStart + i - leftBorder];
}

states[index] = sigmoid(sum);
ready[index] = true;
}
}
__syncthreads();

if (tid == 0) {
int total = counter[0];
for (int i = total; i < total + blockDim.x && i < dim; i++) {
if (ready[i]) {
counter[0]++;
}
}
}
__syncthreads();
}

int n = tid;
while(n < dim) {
outStates[n] = states[n];
n += blockDim.x;
}
}