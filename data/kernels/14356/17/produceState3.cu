#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
return 1 / (1 + expf(-x));
}
__global__ void produceState3(const float* arguments, const int argsSize, const float* weights, const int* topology, const int topSize, float* outStates) {
const int tid = threadIdx.x;
const int dim = argsSize + topSize;
extern __shared__ float s[];
float* states = s;
int* iters = (int*)&states[dim];

if (tid < argsSize) {
states[tid] = arguments[tid];
iters[tid] = 1;
} else {
iters[tid] = 0;
}
__syncthreads();

while(iters[tid] * blockDim.x + tid < dim) {
const int index = iters[tid] * blockDim.x + tid;
const int topIndex = index - argsSize;
const int leftBorder = topology[topIndex*3];
const int rightBorder = topology[topIndex*3 + 1];
const int weightsStart = topology[topIndex*3 + 2];

bool canStart = true;
for (int i = leftBorder; i < rightBorder; i++) {
int threadId = i % blockDim.x;
int mustCounted = i / blockDim.x + 1;
if (iters[threadId] < mustCounted) {
canStart = false;
break;
}
}

if (canStart) {
float sum = 0;
for (int i = leftBorder; i < rightBorder; i++) {
sum += states[i] * weights[weightsStart + i - leftBorder];
}
states[index] = sigmoid(sum);
iters[tid]++;
}
__syncthreads();
}

__syncthreads();

int n = tid;
while(n < dim) {
outStates[n] = states[n];
n += blockDim.x;
}
}