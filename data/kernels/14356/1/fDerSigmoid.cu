#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"
__global__ void fDerSigmoid( const float* arguments, float* results, const long size ) {
const int X = gridDim.x;
const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

if(index < size) {
const float argument = arguments[index];
results[index] = argument - argument * argument;
}
}