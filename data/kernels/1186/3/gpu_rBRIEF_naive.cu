#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ int conflict_free_index(int local_id, int real_idx)
{
return real_idx * 128 + local_id;
}
__global__ void gpu_rBRIEF_naive(float4* workload, int* output, int4* pattern, int4* train_bin_vec, int K, int P, int I)
{
// 0) Memory Setup
extern __shared__ float shared_patchBank[];
int* minVal;
int4   private_pattern[32];
int4   thisBuff, nextBuff;
int train_vec_x, train_vec_y, train_vec_z, train_vec_w;

// coordinate initialize in Private Registers
int coord[96] = { -0, -0, -0, -0, -0, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0, 1, 2, 3, 4,
-0, -4, -3, -2, -1, 0};

// 1) Setup thread ids
int local_id = threadIdx.x;
//int global_id = blockIdx.x * gridDim.x + local_id;

// 2) Load Sampling Pattern into Private Registers
#pragma unroll
for (int i = 0; i < 32; i++)
private_pattern[i] = pattern[i];

// 3) Load my patch into dedicated bank
for (int img = blockIdx.x; img < I; img+=gridDim.x) {

float4* patches;
int * res;
patches = &(workload[img * 3072]); // 128 patches of 24 float4 each
res     = &(output[img * 128]);    // 128 binary vector per image
#ifdef rBRIEFDEBUG
if (threadIdx.x == 0)
printf("Working on img: %d\n", img);
#endif

float4 thisNum;//= make_float4(0.0,0.0,0.0,0.0);
#pragma unroll
for (int i = 1; i < 24; i++) {
thisNum = patches[i * 128 + local_id];
shared_patchBank[conflict_free_index(local_id, i*4 + 0)] = thisNum.x;
shared_patchBank[conflict_free_index(local_id, i*4 + 1)] = thisNum.y;
shared_patchBank[conflict_free_index(local_id, i*4 + 2)] = thisNum.z;
shared_patchBank[conflict_free_index(local_id, i*4 + 3)] = thisNum.w;
}

#ifdef rBRIEFDEBUG
if (threadIdx.x == 0)
printf("Patch is loaded into private registers\n");
#endif

// 4) 1 thread works on 1 patch at a time
float m01 = 0.0;
float m10 = 0.0;
float intensity;
float theta;
#pragma unroll
for (int i = 5; i < 96; i++) {
intensity = shared_patchBank[conflict_free_index(local_id, i)];
m01       = __fmaf_rd(coord[i / 10], intensity, m01);
m10       = __fmaf_rd(coord[i], intensity, m10);
}

//>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
theta = atan2f(m01, m10); // BOTTLE NECK
//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

#ifdef rBRIEFDEBUG
if (threadIdx.x == 0)
printf("m01: %f m10: %f theta: %f\n", m01, m10, theta);
#endif

// 5) Calculate the sin and cos of theta
float sin, cos;
sincosf(theta, &sin, &cos);
#ifdef rBRIEFDEBUG
if (threadIdx.x == 0)
printf("sin: %f cos: %f\n",sin, cos);
#endif

// 6) Sample the patch and return its binary vector
float Ia, Ib;
int ax, ay, bx, by;
unsigned int idxa, idxb;
int rotated_ax, rotated_ay, rotated_bx, rotated_by;
unsigned int binVector = 0;
int result;
#pragma unroll
for (int i = 0; i < 32; ++i) {
ax = private_pattern[i].x;
ay = private_pattern[i].y;
bx = private_pattern[i].z;
by = private_pattern[i].w;

rotated_ax = (int) (cos * ax - sin * ay);
rotated_ay = (int) (-10 * (sin * ay + cos * ay));
rotated_bx = (int) (cos * bx - sin * by);
rotated_by = (int) (-10 * (sin * by + cos * by));

idxa = __sad(rotated_ax, rotated_ay, 0) % 96;
idxb = __sad(rotated_bx, rotated_by, 0) % 96;

Ia = shared_patchBank[conflict_free_index(local_id, idxa)];
Ib = shared_patchBank[conflict_free_index(local_id, idxb)];

result = ((int) Ia > Ib) << i;
binVector |= result;
}

#ifdef rBRIEFDEBUG
if (threadIdx.x == 0) {
printf("%d", binVector);
printf("My Binary vector is: ");
while (binVector) {
if (binVector & 1)
printf("1");
else
printf("0");

binVector >>= 1;
}
printf("\n");
}
#endif

// 7) Preload binary vector from Global Memory and perform Hamming distance calculation
nextBuff = train_bin_vec[0];
float tmp = shared_patchBank[0]; // Borrow one value of shared memory
minVal = (int*) &(shared_patchBank[0]);
for (int i = 1; i < 32; i++) {
thisBuff = nextBuff;
nextBuff = train_bin_vec[i];

*minVal = 32;
train_vec_x = thisBuff.x;
train_vec_x ^= binVector;
train_vec_x = __popc(train_vec_x);
atomicMin(minVal, train_vec_x);
if(train_vec_x == *minVal)
res[i*4 + 0] = local_id;
__syncthreads();

*minVal = 32;
train_vec_y = thisBuff.y;
train_vec_y ^= binVector;
train_vec_y = __popc(train_vec_y);
atomicMin(minVal, train_vec_y);
if(train_vec_y == *minVal)
res[i*4 + 1] = local_id;
__syncthreads();

*minVal = 32;
train_vec_z = thisBuff.z;
train_vec_z ^= binVector;
train_vec_z = __popc(train_vec_z);
atomicMin(minVal, train_vec_z);
if(train_vec_z == *minVal)
res[i*4 + 2] = local_id;
__syncthreads();

*minVal = 32;
train_vec_w = thisBuff.w;
train_vec_w ^= binVector;
train_vec_w = __popc(train_vec_w);
atomicMin(minVal, train_vec_w);
if(train_vec_w == *minVal)
res[i*4 + 3] = local_id;
__syncthreads();
}
shared_patchBank[0] = tmp; // return the shared memory back to normal
}
}