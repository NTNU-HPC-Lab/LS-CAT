#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_rBRIEF_Loop(int N, float4* patches, int4* pattern)
{
// // 1) Shared memory management
// extern __shared__ float4 shared[];
// int4* sharedPattern = (int4*) shared;
// float4* sharedPatches0 = (float4*) &shared[256];
// float4* sharedPatches1 = (float4*) &shared[N*blockDim.x*24 + 256];
// float4* thisPatches;
// float4* nextPatches;
// float4* tmp;
//
// // 2) Load pattern into shared memory (static part of kernel)
// int id = threadIdx.x;
// int stride = blockDim.x;
// for (int i = id; i < 256; i+= stride) {
//   sharedPattern[i] = pattern[i];
// }
//
// // 3) Preload patches 0 into shared memory
// int start = blockIdx.x * (N*24) + id;
// int end   = blockIdx.x * (N*24) + N*24;
// for (int i = start; i < end; i+=stride) {
//   sharedPatches0[i] = patches[i];
// }
// thisPatches = sharedPatches0;

// Kernel Loop begin:
//for (int i = blockIdx.x; i < (P - 1) * N * blockDim.x*24; i+= )

};