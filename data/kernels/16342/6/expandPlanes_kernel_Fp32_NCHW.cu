#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
// Block size of 256, same mask/val for 64 consecutive threads.
constexpr int kNumShmemElements = 256 / 64;

__shared__ uint64_t shMasks[kNumShmemElements];
__shared__ float shVals[kNumShmemElements];

int index = threadIdx.x + blockDim.x * blockIdx.x;

int planeIndex = index >> 6;

if (planeIndex >= n) return;

// Load inputs to shared memory.
if (threadIdx.x < kNumShmemElements) {
shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
}
__syncthreads();

uint64_t mask = shMasks[threadIdx.x >> 6];

int sqIndex = index & 0x3F;
float op = 0;

bool set = !!(mask & (1ull << sqIndex));
if (set) {
op = shVals[threadIdx.x >> 6];
}
output[index] = op;
}