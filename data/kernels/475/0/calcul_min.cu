#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcul_min( unsigned long *ord, int ind_start, int ind_end, unsigned long long *ymin, int *ind_min, int size_max_parallel ){

int a = threadIdx.x;
int size_tot = (ind_end - ind_start -1);

//On n'effectue pas le calcul aux indices ind_start ni ind_end
int nb_threads = ceilf((float)size_tot/(float)size_max_parallel);

//size of region to compute in the current thread
int size_parallel = ceilf( (float)size_tot/(float)nb_threads );


//have to be computed before the case of a different size_parallel value
int ind_start_loc = ind_start + a * size_parallel + 1;

if ( a == (nb_threads - 1) )
size_parallel = size_tot - (nb_threads - 1) * size_parallel;


unsigned long min_loc = ord[ind_start_loc];
int ind_min_loc = ind_start_loc;
int i = 0;

//printf("FINDING YMIN\n");

for ( i = ind_start_loc; i < ind_start_loc + size_parallel; i++ ){

//Looking for the lowest ordinate
if ( ord[i]< min_loc ){
min_loc = ord[i];
ind_min_loc = i;

}

}

atomicMin(ymin, min_loc);

__syncthreads();

if (*ymin == min_loc)
*ind_min = ind_min_loc;

return;
}