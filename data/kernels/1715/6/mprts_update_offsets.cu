#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void mprts_update_offsets(int nr_total_blocks, uint *d_off, uint *d_spine_sums)
{
int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

if (bid <= nr_total_blocks) {
d_off[bid] = d_spine_sums[bid * CUDA_BND_STRIDE + 0];
}
}