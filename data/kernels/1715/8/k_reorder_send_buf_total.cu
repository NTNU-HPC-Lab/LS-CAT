#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint *d_bidx, uint *d_sums, float4 *d_xi4, float4 *d_pxi4, float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
if (i >= nr_prts)
return;

if (d_bidx[i] == CUDA_BND_S_OOB) {
int j = d_sums[i];
d_xchg_xi4[j]  = d_xi4[i];
d_xchg_pxi4[j] = d_pxi4[i];
}
}