#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient_descent ( float * d_k, float * o_i, float * g_ik, unsigned int size_d )
{
// X = Node Delta Count (layer k)
int x = blockIdx.x * blockDim.x + threadIdx.x;
// Y = Node Output Count (layer i)
int y = blockIdx.y * blockDim.y + threadIdx.y;
// Row-Major Matrix
g_ik[size_d*x+y] = __fmul_rz( d_k[x], o_i[y]);
}