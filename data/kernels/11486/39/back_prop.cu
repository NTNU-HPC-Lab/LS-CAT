#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back_prop ( float * weight, float * gradient, float * update, float alpha, float epsilon )
{
// X Grid iterates weight, gradient and update (all same size)
int x = blockIdx.x * blockDim.x + threadIdx.x;
// ε * ( ∂E / ∂W[ik] )
float lhs = __fmul_rz( epsilon, gradient[x] );
// α * ( Δw(t-1) )
float rhs = __fmul_rz( alpha, update[x] );
// Δw(t) = ε * ( ∂E / ∂W[i] ) + α * ( Δw(t-1) )
float d_w = __fadd_rz( lhs, rhs );

//printf("Δw(t): %f W[i]: %f W[i]+Δw(t): %f Δw(t-1): %f\n",d_w,weight[x],__fadd_rz(weight[x],d_w),update[x]);

// Update weight: W[i] = W[i] + Δw(t)
weight[x] = __fadd_rz( weight[x], d_w );
// Set `Δw(t-1) = Δw(t)`
update[x] = d_w;
}