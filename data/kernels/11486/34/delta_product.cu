#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void delta_product ( const float * w_ik, const float * d_k, float * output, unsigned int width )
{
// X is layer[i] nodes (size_i)
int x = blockIdx.x * blockDim.x + threadIdx.x;
// Y is layer[k] nodes (size_k) == d_k == w_per_n
int y = blockIdx.y * blockDim.y + threadIdx.y;
//  W[ik] * δ[k] - Row-Major Matrix
output[width*x+y] = __fmul_rz( d_k[y], w_ik[width*x+y]);
}