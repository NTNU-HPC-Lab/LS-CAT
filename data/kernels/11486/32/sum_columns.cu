#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_columns ( float * w_mtx, float * output, unsigned int height, unsigned int width )
{
// X thread iterates Columns and sums their Row values
int x = blockIdx.x * blockDim.x + threadIdx.x;
float total;
for ( int y = 0; y < height; y++ )
{
total = __fadd_rz( total, w_mtx[y*width+x]);
}
output[x] = total;
}