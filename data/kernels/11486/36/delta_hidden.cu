#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void delta_hidden ( float * prime_ji, float * delta_i )
{
// X grid is size_i
int x = blockIdx.x * blockDim.x + threadIdx.x;
// δ[i] = f'( Σ[ji]) * Σ(w[ik] * δ[k])
// NOTE: delta_i ALREADY contains `Σ(w[ik] * δ[k])`
float rhs = delta_i[x];
// δ[i] = σ'( Σ[ji]) * Σ(w[ik] * δ[k])
delta_i[x] = __fmul_rz( prime_ji[x], rhs );
}