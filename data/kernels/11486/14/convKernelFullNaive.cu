#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convKernelFullNaive(float* d_Input, float* d_Output, float* d_Kernel, int  imageW, int imageH, int kernelR)
{
int row = blockDim.y * blockIdx.y + threadIdx.y;
int col = blockDim.x * blockIdx.x + threadIdx.x;

int loc = row * imageW + col;

float s = 0;
float t = 0;

for (int i = -KERNAL_RAD; i <= KERNAL_RAD; i++)
for (int j = -KERNAL_RAD; j <= KERNAL_RAD; j++)
{
t = 0;

if (row  + i >= 0 && row  + i < imageH && col  + j >= 0 && col  + j < imageW )
t = d_Input[loc + i * imageW + j];

s += t * d_Kernel[(KERNAL_RAD - i) * (KERNAL_RAD + KERNAL_RAD + 1) + KERNAL_RAD - j];
}
d_Output[loc] = s;
}