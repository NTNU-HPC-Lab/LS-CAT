#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_gradients ( float * gradient, float * new_value )
{
// X Grid iterates all gradient values
int x = blockIdx.x * blockDim.x + threadIdx.x;
// A Simple summation
gradient[x] = __fadd_rz( gradient[x], new_value[x] );
}