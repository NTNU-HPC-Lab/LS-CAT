#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squared_error ( const float * ideal, float * actual, float * errors )
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
float diff = ideal[x] - actual[x];
errors[x] = __fmul_rz(diff,diff);
//printf("squared_error: %f, ideal: %f, actual: %f\n",errors[x],ideal[x],actual[x]);
}