#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void delta_sum_rows ( float * w_ik_d, float * delta_i, unsigned int width )
{
// X thread iterates Rows and Sums the respective Column values
int x = blockIdx.x * blockDim.x + threadIdx.x;
float total = 0.f;
for ( int y = 0; y < width; y++ )
{
//printf("X:%d, Σ: %.9f + %.9f\n",x,total,w_ik_d[x*width+y]);
total = __fadd_rz( total, w_ik_d[x*width+y]);
}
delta_i[x] = total;
}