#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void delta_output ( const float * primed_sum, const float * ideal, const float * actual, float * delta, unsigned int index )
{
// x is the output neuron/node count (e.g., length of actual & ideal)
int x = blockIdx.x * blockDim.x + threadIdx.x;

// Calculate the Negative Error: -(Actual - Ideal)
float neg_error = __fmul_rz(-1,(actual[x] - ideal[x]));

// -E * σ'(Σ(O[i])
delta[x+index] = __fmul_rz( neg_error, primed_sum[x+index] );
}