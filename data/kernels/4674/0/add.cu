#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// function generate random numbers and assign it to array
__global__ void add(int *a, int *b, int *c) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}