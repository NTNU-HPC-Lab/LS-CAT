#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecReduce(float* g_idata, float* g_odata, int N)
{
// shared memory size declared at kernel launch
extern __shared__ float sdata[];

unsigned int tid = threadIdx.x;
unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;

// For thread ids greater than data space
if (globalid < N) {
sdata[tid] = g_idata[globalid];
}
else {
sdata[tid] = 0;  // Case of extra threads above N
}

// each thread loads one element from global to shared mem
__syncthreads();

// do reduction in shared mem
for (unsigned int s=blockDim.x / 2; s > 0; s = s >> 1) {
if (tid < s) {
sdata[tid] = sdata[tid] + sdata[tid+ s];
}
__syncthreads();
}

// write result for this block to global mem
if (tid == 0)  {
g_odata[blockIdx.x] = sdata[0];
}
}