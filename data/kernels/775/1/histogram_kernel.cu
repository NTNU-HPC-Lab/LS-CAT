#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_kernel(int* PartialHist, int* DeviceData, int DataCount,int* timer)
{
int tid = threadIdx.x;
int gid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
clock_t start_atomic=0;
clock_t stop_atomic=0;

extern __shared__ int hist[];

if(tid==0)
{
start_atomic = clock();
}

for(int i = 0; i< H; i++)
hist[tid * H + i] = 0;

for(int j = gid; j < DataCount; j += stride)
hist[tid * H + DeviceData[j]]++;

__syncthreads();

for(int t_hist = 0; t_hist < blockDim.x; t_hist++)
{
atomicAdd(&PartialHist[tid],hist[t_hist * H + tid]);
atomicAdd(&PartialHist[tid + blockDim.x],hist[t_hist * H + tid + blockDim.x]);
}
stop_atomic=clock();

if(tid==0)
{
timer[blockIdx.x] = stop_atomic - start_atomic;
}
}