#include "hip/hip_runtime.h"
#include "includes.h"

#define H 	64

// Default values
int N = 10000; 		//Size
int T = 32; 		//BlockSize
int B = 4; 		//Blocks

// Host Variables
int* HostData;
int* HostHist;
int* HostTimer=NULL;

// Device Variables
int* DeviceData;
int* DeviceHist;
int* DeviceTimer=NULL;

// Timer Variables
struct timeval CPU_Time_start, CPU_Time_end;
struct timeval GPU_Time_start, GPU_Time_end;
struct timeval DeviceToHost_start, DeviceToHost_end;
struct timeval HostToDevice_start, HostToDevice_end;
struct timeval CPU_Partial_Time_start, CPU_Partial_Time_end;
struct timeval CPU_Cleanup_Time_start, CPU_Cleanup_Time_end;
struct timeval Total_Time_start, Total_Time_end;


// Function Declaration
void Cleanup(void);
void HistogramSequential(int* result, int* data, int size);

// Histogram kernel


__global__ void histogram_kernel(int* PartialHist, int* DeviceData, int dataCount,int* timer)
{
unsigned int tid = threadIdx.x;
unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int stride = blockDim.x * gridDim.x;
clock_t start_clock=0;
clock_t stop_clock=0;

if(tid==0)
{
start_clock = clock();
}

__shared__ int BlockHist[H];

extern __shared__ int hist[];

for(int h = 0; h < H; h++)
{
hist[tid * H + h]=0;
}

BlockHist[tid] = 0;
BlockHist[tid + blockDim.x] = 0;

for(int pos = gid; pos < dataCount; pos += stride)
hist[tid * H + DeviceData[pos]]++;

for(int t_hist = 0; t_hist < blockDim.x; t_hist++)
{
BlockHist[tid] += hist[t_hist * H + tid];
BlockHist[tid+blockDim.x] += hist[(t_hist * H)+(tid + blockDim.x)];
}

PartialHist[tid+(blockIdx.x * H)] = BlockHist[tid];
PartialHist[tid+(blockIdx.x * H) + blockDim.x] = BlockHist[tid + blockDim.x];

if(tid==0)
{
stop_clock = clock();
timer[blockIdx.x * 2] = start_clock;
timer[blockIdx.x * 2 + 1] = stop_clock;
}
}