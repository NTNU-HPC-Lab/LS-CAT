#include "hip/hip_runtime.h"
#include "includes.h"


#define DATA float
#define BOOL int
#define MAX_ERR (float)1e-5
#define MAX_EPOCHS 3

#define MAX(a,b) ((a)>(b)?(a):(b))
#define MIN(a,b) ((a)<(b)?(a):(b))

//Grid features
//Leggere 15 febbraio del diario (passo 1 del feedforward, considerazioni)

#define OPTIMUM_BLOCK_NUM 4 //In vista della concorrenza dei kernels
#define BLOCK_SIDE	16

#define OPTIMUM_BLOCK_NUM_FIRST_LAYER 2
#define BLOCK_SIDE_FIRST_LAYER 32

/*Struct Grid Settings*/

typedef struct grid_settings {
int grid[3];
int block[3];
}grid_settings;

grid_settings gs = { { OPTIMUM_BLOCK_NUM_FIRST_LAYER, OPTIMUM_BLOCK_NUM, OPTIMUM_BLOCK_NUM },{ BLOCK_SIDE_FIRST_LAYER,BLOCK_SIDE,BLOCK_SIDE } };

//Network features

#define NEURO_INPUT 784 //#neuroni dell'input layer
#define NEURO_H_0	56	//#neuroni del primo hidden layer
#define NEURO_H_1	28	//#neuroni del secondo hidden layer
#define NEURO_OUTPUT 10 //#neuroni dell'output layer
#define TOTAL_PATT	60000 //#patterns totali
#define NUM_HIDDEN 2 //#hidden layers
#define TOTAL_LAYER 4 //#di layers

//Streams Settings
#define NSTREAMS 3

//Texture reference (FOR TARGET MATRIX)
texture<DATA, 2, hipReadModeElementType> texreference_target;

//Constant memory (read by all the threads)
__constant__ DATA alpha_const[1];
__constant__ DATA eta_const[1];

/*UTILITIES*/

__global__ void updateWeightBiasMat(DATA *delta_weightbias, DATA *weight, int rows, int cols) {

int dest_x = blockIdx.x*blockDim.x + threadIdx.x;
int dest_y = blockIdx.y*blockDim.y + threadIdx.y;

if (dest_x < cols && dest_y < rows) {
DATA derivative = delta_weightbias[dest_y*cols + dest_x];
weight[dest_y*cols + dest_x] += derivative;
delta_weightbias[dest_y*cols + dest_x] *= alpha_const[0];
}
}