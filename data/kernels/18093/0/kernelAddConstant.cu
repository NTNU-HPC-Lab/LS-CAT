#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDAMAXTHREADPERBLOCK 1024
#define CUDAMAXBLOCK 65536

using namespace std;

__global__ void kernelAddConstant(int *g_a, const int b)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
g_a[idx] += b;
}