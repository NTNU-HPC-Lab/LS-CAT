#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE  32
#define N           3200



__global__ void matMult(float* a, float* b, int n, float* c)
{

int   bx = blockIdx.x;
int   by = blockIdx.y;
int   tx = threadIdx.x;
int   ty = threadIdx.y;
float sum = 0.0f;
int   ia = n * BLOCK_SIZE * by + n * ty;
int   ib = BLOCK_SIZE * bx + tx;


for (int k = 0; k < n; k++)
sum += a[ia + k] * b[ib + k * n];

int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

c[ic + n * ty + tx] = sum;
}