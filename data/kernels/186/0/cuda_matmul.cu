#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_matmul(float* A, float* B, float* C, size_t n)
{
float sum = 0.0f;

#ifndef MATMUL_USE_SHARED
int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
int ib = blockDim.x * blockIdx.x + threadIdx.x;
int ic = ia + ib;

// Multiply two matrices
for (int k = 0; k < n; k++)
sum += A [ia + k] * B [ib + k * n];
#else
// Base indexes inside A and B
int ia = (blockDim.y * blockIdx.y) * n;
int ib = blockDim.x * blockIdx.x;

// Subindex inside a "tile"
int tileidx = n * threadIdx.y + threadIdx.x;

// Index in C
int ic = ia + ib + tileidx;

int aoff = 0, boff = 0;

// Shared memory for the "tile" sub-matrix of A and B
__shared__ float As [BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs [BLOCK_SIZE][BLOCK_SIZE];

// Go through "tiles" of size blockDim.x * blockDim.y
for (; aoff < n; aoff += blockDim.x, boff += blockDim.y * n)
{
// Load the "tile" matrices from global memory to shared memory
As [threadIdx.y][threadIdx.x] = A [ia + aoff + tileidx];
Bs [threadIdx.y][threadIdx.x] = B [ib + boff + tileidx];

// Synchronize to make sure the matrices are loaded
__syncthreads();

// Multiply the two matrices
for (int k = 0; k < BLOCK_SIZE; k++)
sum += As [threadIdx.y][k] * Bs [k][threadIdx.x];

// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
__syncthreads();
}
#endif
// Write the block sub-matrix to global memory
// each thread writes one element
C [ic] = sum;
}