#include "hip/hip_runtime.h"
#include "includes.h"



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = a[i] + b[i];
i += blockDim.x * gridDim.x;
}