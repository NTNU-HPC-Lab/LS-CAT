#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Alexandre Maros - 2016
*
* Cuda Matrix Multiplication with Global Memory.
*
* nvcc cuda_matrix_global.cu -o cg.o
*
* Implemented by Alexandre Maros for learning purposes.
* A version of this code using Shared Memory is in here:
* https://github.com/alepmaros/cuda_matrix_multiplication
*
* Distributed under the MIT Lincese.
*/


//32x32
#define NTHREADS_X 16
#define NTHREADS_Y 32
#define THREADS_PER_BLOCK NTHREADS_X * NTHREADS_Y

/* A macro used for error checking in CUDA function calls
* Credit to: http://stackoverflow.com/a/14038590 for the gpuErrchk macro.
*/
__global__ void matrix_mul(int *a, int *b, int *c, int a_ncolumns, int c_nlines, int c_ncolumns)
{

int column = blockIdx.x * blockDim.x + threadIdx.x;
int line =  blockIdx.y * blockDim.y + threadIdx.y;

if (column  >= c_ncolumns || line >= c_nlines)
return;

int i, sum = 0;


int beginA = a_ncolumns * line;
int beginB = column;

for (i = 0; i < a_ncolumns; i++)
{
sum += a[beginA + i] * b[i * c_ncolumns + beginB];
}

c[line * c_ncolumns + column] = sum;
}