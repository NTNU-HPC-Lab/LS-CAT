#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataSubtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
CUDA_KERNEL_LOOP(index, nthreads) {
int x = index % topwidth; //w-pos
int y = (index / topwidth) % topheight; //h-pos
int c = (index / topwidth / topheight) % topchannels; //channels

// Offset of patch in image 2
int s2o = (c % neighborhood_grid_width + x_shift) * stride2;

// First (upper left) position of kernel center in current neighborhood in image 1
int x1 = x*stride1 + kernel_radius + max_displacement;
int y1 = y*stride1 + kernel_radius + 0;

// Iterate through 3D patch
float sum = 0;
for(int j = -kernel_radius; j <= kernel_radius; j++) { // HEIGHT
for(int i = -kernel_radius; i <= kernel_radius; i++) { // WIDTH
for(int l = 0; l < bottomchannels; l++) { // CHANNELS
// Calculate position in image 2
int x2 = x1 + s2o;
int y2 = y1;

// Indices in bottom data: (CH=l,W=x2,H=y2,N)
int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + l;
int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + l;

// Do the correlation:
sum += fabsf(bottom0[idx1] - bottom1[idx2]);
}
}
}
const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
top[index + item*topcount] = sum / (float)sumelems;
}

}