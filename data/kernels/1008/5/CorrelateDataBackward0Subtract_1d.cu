#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward0Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, float *bottom0diff, const float *bottom0, const float *bottom1, const float *topdiff)
{
CUDA_KERNEL_LOOP(index, nthreads) {
int l = index % bottomwidth + pad_size; //w-pos
int m = (index / bottomwidth) % bottomheight; //h-pos
int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels

//Get X,Y ranges and clamp
// round_off is a trick to enable integer division with ceil, even for negative numbers
// We use a large offset, for the inner part not to become negative.
const int round_off = ROUND_OFF;
const int round_off_s1 = stride1 * round_off;

// We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
int ymin = (m - 2*kernel_radius - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1

// Same here:
int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
int ymax = (m - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1


float sum = 0;
if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
{
xmin = max(0,xmin);
xmax = min(topwidth-1,xmax);

ymin = max(0,ymin);
ymax = min(topheight-1,ymax);

{
for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

// Get bottom1 data:
int s2o = stride2 * o;
int idxbot = ((item * pbottomheight + (m)) * pbottomwidth + (l+s2o)) * bottomchannels + n;
float bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m,n]
float bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m,n]
float sign = (bot0tmp >= bot1tmp) ? float(1.0) : float(-1.0);

// Index offset for topdiff in following loops:
int op = (o-x_shift); // index [o,p]
int idxopoffset = (item * topchannels + op);

for(int y = ymin; y <= ymax; y++) {
for(int x = xmin; x <= xmax; x++) {
int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
sum += topdiff[idxtopdiff] * sign;
}
}
}
}
}
const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
bottom0diff[index + item*bottomcount] = sum / (float)sumelems;
}

}