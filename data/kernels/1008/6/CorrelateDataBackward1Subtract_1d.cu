#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward1Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, const float *bottom0, const float *bottom1, float *bottom1diff, const float *topdiff)
{
CUDA_KERNEL_LOOP(index, nthreads) {
int l = index % bottomwidth + pad_size; //w-pos
int m = (index / bottomwidth) % bottomheight; //h-pos
int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels

// round_off is a trick to enable integer division with ceil, even for negative numbers
// We use a large offset, for the inner part not to become negative.
const int round_off = ROUND_OFF;
const int round_off_s1 = stride1 * round_off;

float sum = 0;
{
for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

int s2o = stride2 * o;

//Get X,Y ranges and clamp
// We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
int ymin = (m - 2*kernel_radius - 0 - 0 + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1

// Same here:
int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
int ymax = (m - 0 - 0 + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - s2p) / stride1

if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
{
xmin = max(0,xmin);
xmax = min(topwidth-1,xmax);

ymin = max(0,ymin);
ymax = min(topheight-1,ymax);

// Get bottom0 data:
int idxbot = ((item * pbottomheight + (m)) * pbottomwidth + (l-s2o)) * bottomchannels + n;
float bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m,n]
float bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m,n]
float sign = (bot0tmp >= bot1tmp) ? float(-1.0) : float(1.0);

// Index offset for topdiff in following loops:
int op = (o-x_shift); // index [o,p]
int idxOpOffset = (item * topchannels + op);

for(int y = ymin; y <= ymax; y++) {
for(int x = xmin; x <= xmax; x++) {
int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
sum += topdiff[idxtopdiff] * sign;
}
}
}
}
}
const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
bottom1diff[index + item*bottomcount] = sum / (float)sumelems;
}

}