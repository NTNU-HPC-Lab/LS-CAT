#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobel( int width_d, int height_d, int threshold_d, unsigned int *pic_d , int *final_res)
{


int row_1 = blockIdx.y * blockDim.y + threadIdx.y;

int col_1 = blockIdx.x * blockDim.x + threadIdx.x;

int tx = threadIdx.y;

int ty = threadIdx.x;

int width_Tile = TILE_SIZE;

int id, id1;

__shared__ int sharedTile[TILE_SIZE * TILE_SIZE];

int magnitude, sum1, sum2;

// Shared Tile Initialization
sharedTile[tx * width_Tile + ty]  = 0;

__syncthreads();

// Copying Data from Global to Shared Memory
sharedTile[tx * width_Tile + ty] = pic_d[row_1 * (width_d) + col_1];

__syncthreads();

// Output
if ((row_1 < height_d) && (col_1 < width_d))
{

final_res[row_1 * width_d + col_1] = 0;

}

__syncthreads();


if (row_1 > 0 && col_1 > 0 && row_1 < height_d - 1 && col_1 < width_d - 1)
{
// Applying Sobel Filter on the Tile Stored in the Shared Memory
if ((tx > 0) && (tx < width_Tile - 1)  && (ty > 0) && (ty < width_Tile - 1))
{
id = row_1 * width_d + col_1;

sum1 = sharedTile[ width_Tile * (tx-1) + ty+1] - sharedTile[ width_Tile * (tx-1) + ty-1 ] + 2 * sharedTile[ width_Tile * (tx)   + ty+1 ] - 2 * sharedTile[ width_Tile*(tx)   + ty-1 ] +  sharedTile[ width_Tile * (tx+1) + ty+1] - sharedTile[ width_Tile*(tx+1) + ty-1 ];

sum2 = sharedTile[ width_Tile * (tx-1) + ty-1 ] + 2 * sharedTile[ width_Tile * (tx-1) + ty ] + sharedTile[ width_Tile * (tx-1) + ty+1] - sharedTile[width_Tile * (tx+1) + ty-1 ] - 2 * sharedTile[ width_Tile * (tx+1) + ty ] - sharedTile[ width_Tile * (tx+1) + ty+1];

magnitude = sum1 * sum1 + sum2 * sum2;

if (magnitude > threshold_d)

{

final_res[id] = 255;

}

else

{

final_res[id] = 0;

}

}

__syncthreads();

// For the Pixels at the Boundaries of the Block using Global Memory

if ((row_1 == blockIdx.y * blockDim.y + blockDim.y - 1) || (col_1 == blockIdx.x * blockDim.x + blockDim.x - 1) || (row_1 == blockIdx.y * blockDim.y) || (col_1 == blockIdx.x * blockDim.x))

{
id1 = row_1 * width_d + col_1;

sum1 =  pic_d[ width_d * (row_1-1) + col_1+1] - pic_d[ width_d * (row_1-1) + col_1-1 ] + 2 * pic_d[ width_d * (row_1) + col_1+1 ] - 2 * pic_d[ width_d*(row_1)   + col_1-1 ] + pic_d[ width_d * (row_1+1) + col_1+1] - pic_d[ width_d*(row_1+1) + col_1-1 ];

sum2 = pic_d[ width_d * (row_1-1) + col_1-1 ] + 2 * pic_d[ width_d * (row_1-1) + col_1 ] + pic_d[ width_d * (row_1-1) + col_1+1] - pic_d[width_d * (row_1+1) + col_1-1 ] - 2 * pic_d[ width_d * (row_1+1) + col_1 ] - pic_d[ width_d * (row_1+1) + col_1+1];



magnitude =  sum1*sum1 + sum2*sum2;

if (magnitude > threshold_d)

{

final_res[id1] = 255;

}

else

{

final_res[id1] = 0;

}

}

__syncthreads();

}

}