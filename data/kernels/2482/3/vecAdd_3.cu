#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_3(double *a, double *b, double *c, int n)
{
int id = threadIdx.x;
int id_1;

for(int i = 0; i < n; i++)
{
id_1 = id + i * n;

c[id_1] = a[id_1] + b[id_1];

}

}