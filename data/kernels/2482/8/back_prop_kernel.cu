#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back_prop_kernel(float *device_output, float *inP, float *m_hidden, float* weights_2, float* o_errG, int nInput, int nHidden, int nOutput,  float l_R)
{
int linearThreadIndex = threadIdx.x;

int unit = blockIdx.x;

__shared__ float weightedSum[1];

if (linearThreadIndex==0)
{
for (int i=0; i<nOutput; i++)
{

weightedSum[0] += weights_2[unit*nOutput + i] * o_errG[i];

}

}

__syncthreads();

if (linearThreadIndex < nInput)
{

device_output[linearThreadIndex*nHidden + unit] = l_R * inP[linearThreadIndex] * m_hidden[unit]*(1 - m_hidden[unit]) * weightedSum[0];

}

}