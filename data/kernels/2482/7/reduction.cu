#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(float *g_data, int n)
{
__shared__ float s_data[NUM_ELEMENTS];

int tid = threadIdx.x;

int myIndex = threadIdx.x + blockIdx.x*blockDim.x;

//s_data[tid] = 0.0;

s_data[tid] = g_data[myIndex];

__syncthreads();

for(int s = blockDim.x / 2; s > 0; s >>=1)
{
if(tid < s)
{

s_data[tid] += s_data[tid + s];

}

__syncthreads();
}

if (tid == 0)
{

g_data[blockIdx.x] = s_data[tid];

}


}