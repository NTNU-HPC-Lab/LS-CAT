#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_2(double *a, double *b, double *c, int n)
{
int id = threadIdx.x;
int id_1;

for(int i = 0; i < n; i++)
{
id_1 = id * n + i;

c[id_1] = a[id_1] + b[id_1];

}
}