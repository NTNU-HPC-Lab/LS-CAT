#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back_prop_kernel_batch(float *device_output, float *inP, float *m_hidden, float* weights_2, float* o_errG, int nInput, int nHidden, int nOutput, float l_R, int batchSize)
{
int linearThreadIndex = threadIdx.x;

int unit = blockIdx.x%nHidden;

int batch = blockIdx.x/nHidden;

__shared__ float weightedSum[1];

float temp = 0.0;

if (linearThreadIndex ==0 && unit<nHidden)
{
for (int i=0; i<nOutput; i++)
{

weightedSum[0] += weights_2[unit*nOutput + i] * o_errG[batch*(nOutput+1) +i];

}

}

__syncthreads();

if (linearThreadIndex < nInput)
{
temp = l_R * inP[batch*(nInput+1) + linearThreadIndex] * m_hidden[batch*(nHidden+1) + unit]*(1 - m_hidden[batch*(nHidden+1) + unit]) * weightedSum[0];

atomicAdd(&device_output[linearThreadIndex*nHidden + unit], temp);

}


}