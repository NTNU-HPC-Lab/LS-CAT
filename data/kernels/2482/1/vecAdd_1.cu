#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_1(double *a, double *b, double *c, int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < n)
{
c[id] = a[id] + b[id];
}

}