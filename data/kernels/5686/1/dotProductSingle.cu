#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dotProductSingle(int* pFeatureList, float* pValuesList, size_t* pSizeOfInstanceList, size_t pSize, size_t pMaxNnz, float* pDevDotProduct) {
int instanceId = blockIdx.x;
int threadId = threadIdx.x;
float __shared__ value[32];
int __shared__ jumpLength;
size_t __shared__ size;


while (instanceId < pSize) {
value[threadIdx.x] = 0;
if (threadIdx.x == 0) {
jumpLength = instanceId * pMaxNnz;
size = pSizeOfInstanceList[instanceId];
}
__syncthreads();
while (threadId < size) {
value[threadIdx.x] += pValuesList[jumpLength + threadId] *  pValuesList[jumpLength + threadId];

threadId += blockDim.x;
}
// reduce
__syncthreads();
int i = blockDim.x/2;
while (i != 0) {
if (threadIdx.x < i) {
value[threadIdx.x] += value[threadIdx.x + i];
}
__syncthreads();
i /= 2;
}

pDevDotProduct[instanceId] = value[0];
instanceId += gridDim.x;
threadId = threadIdx.x;
}
}