#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void euclideanDistanceCuda(float3* pDotProducts, size_t pSize, float* results) {
int instance = blockIdx.x * blockDim.x + threadIdx.x;

while (instance < pSize) {
results[instance] = pDotProducts[instance].x - 2*pDotProducts[instance].y + pDotProducts[instance].z;
if (results[instance] < 0.0) results[instance] = 0.0;
instance += gridDim.x;
}
}