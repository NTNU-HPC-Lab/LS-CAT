#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeDotProducts(float3* pDotProducts, size_t pSize, int* pCandidates, size_t* pJumpLength, size_t* pCandidateSize, size_t pNumberOfCandidates, int* pFeatureIdsNeighbor, float* pValuesNeighbor, size_t pMaxNnzNeighbor, size_t* pSizeNeighbor, int* pFeatureIdsInstance, float* pValuesInstance, size_t pMaxNnzInstance, size_t* pSizeInstance, float* pPreComputedDotProductsNeighbor, float* pPreComputedDotProductsInstance) {


int instanceCandidates = blockIdx.x;
int round = 0;
int i = 0;
const int threadCount = 32;
__shared__ int instanceCounter;
__shared__ int neighbor;
__shared__ int instance;

__shared__ int featureIdX[threadCount];
__shared__ int featureIdY[threadCount];
__shared__ float value[threadCount];
__shared__ int pStartPosX;
__shared__ int pEndPosX;
__shared__ int pStartPosY;
__shared__ int pEndPosY;

while (instanceCandidates < pNumberOfCandidates) {
if (threadIdx.x == 0) {
neighbor = pCandidates[pJumpLength[instanceCandidates]];
instanceCounter = 0;
}
__syncthreads();
while (instanceCounter < pCandidateSize[neighbor]) {

if (threadIdx.x == 0) {
instance = pCandidates[pJumpLength[instanceCandidates]+instanceCounter];
pStartPosX = neighbor*pMaxNnzNeighbor;
pEndPosX = neighbor*pMaxNnzNeighbor + pSizeNeighbor[neighbor];
pStartPosY = instance*pMaxNnzInstance;
pEndPosY = instance*pMaxNnzInstance + pSizeInstance[instance];
}
value[threadIdx.x] = 0.0;

__syncthreads();

while (pStartPosX < pEndPosX+threadCount - (pEndPosX%threadCount) && pStartPosY < pEndPosY+threadCount - (pEndPosY%threadCount) ) {

featureIdX[threadIdx.x] = pFeatureIdsNeighbor[pStartPosX + threadIdx.x];
featureIdY[threadIdx.x] = pFeatureIdsInstance[pStartPosY + threadIdx.x];

while (round < threadCount) {
if (featureIdX[(threadIdx.x + round) % threadCount] == featureIdY[threadIdx.x]) {
value[threadIdx.x] += pValuesNeighbor[pStartPosX + ((threadIdx.x + round) % threadCount)] * pValuesInstance[pStartPosY + threadIdx.x];
break;
}
++round;
}
__syncthreads();
round = 0;
if (threadIdx.x == 0) {
if (featureIdX[threadCount-1] == featureIdY[threadCount-1]) {
pStartPosY += threadCount;
pStartPosX += threadCount;
} else if (featureIdX[threadCount-1] < featureIdY[threadCount-1]) {
pStartPosX += threadCount;
} else {
pStartPosY += threadCount;
}
}
__syncthreads();
}
__syncthreads();

i = blockDim.x/2;
while (i != 0) {
if (threadIdx.x < i) {
value[threadIdx.x] += value[threadIdx.x + i];
}
__syncthreads();
i /= 2;
}
if (threadIdx.x == 0) {
pDotProducts[pJumpLength[instanceCandidates]+instanceCounter].y = value[0];
pDotProducts[pJumpLength[instanceCandidates]+instanceCounter].x = pPreComputedDotProductsNeighbor[neighbor];
pDotProducts[pJumpLength[instanceCandidates]+instanceCounter].z = pPreComputedDotProductsInstance[instance];
++instanceCounter;
}
__syncthreads();
}
instanceCandidates += gridDim.x;
}
}