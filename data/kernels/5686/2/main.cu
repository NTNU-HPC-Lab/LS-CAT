#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computeDotProducts.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float3 *pDotProducts = NULL;
hipMalloc(&pDotProducts, XSIZE*YSIZE);
size_t pSize = XSIZE*YSIZE;
int *pCandidates = NULL;
hipMalloc(&pCandidates, XSIZE*YSIZE);
size_t *pJumpLength = NULL;
hipMalloc(&pJumpLength, XSIZE*YSIZE);
size_t *pCandidateSize = NULL;
hipMalloc(&pCandidateSize, XSIZE*YSIZE);
size_t pNumberOfCandidates = 1;
int *pFeatureIdsNeighbor = NULL;
hipMalloc(&pFeatureIdsNeighbor, XSIZE*YSIZE);
float *pValuesNeighbor = NULL;
hipMalloc(&pValuesNeighbor, XSIZE*YSIZE);
size_t pMaxNnzNeighbor = 1;
size_t *pSizeNeighbor = NULL;
hipMalloc(&pSizeNeighbor, XSIZE*YSIZE);
int *pFeatureIdsInstance = NULL;
hipMalloc(&pFeatureIdsInstance, XSIZE*YSIZE);
float *pValuesInstance = NULL;
hipMalloc(&pValuesInstance, XSIZE*YSIZE);
size_t pMaxNnzInstance = 1;
size_t *pSizeInstance = NULL;
hipMalloc(&pSizeInstance, XSIZE*YSIZE);
float *pPreComputedDotProductsNeighbor = NULL;
hipMalloc(&pPreComputedDotProductsNeighbor, XSIZE*YSIZE);
float *pPreComputedDotProductsInstance = NULL;
hipMalloc(&pPreComputedDotProductsInstance, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeDotProducts<<<gridBlock,threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computeDotProducts<<<gridBlock,threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computeDotProducts<<<gridBlock,threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}