#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cosineSimilarityCuda(float3* pDotProducts, size_t pSize, float* results) {
int instance = blockIdx.x * blockDim.x + threadIdx.x;

while (instance < pSize) {
results[instance] = pDotProducts[instance].y / (sqrtf(pDotProducts[instance].x)* sqrtf(pDotProducts[instance].z));
instance += gridDim.x;
}
}