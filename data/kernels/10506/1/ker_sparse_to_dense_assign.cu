#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
trg[id] = src[idx[id]];
}