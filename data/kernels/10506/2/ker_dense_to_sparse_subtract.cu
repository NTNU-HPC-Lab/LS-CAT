#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
// Get our global thread ID
int id = blockIdx.x*blockDim.x+threadIdx.x;

// Make sure we do not go out of bounds
if (id < n)
atomicAdd(trg + idx[id], -src[id]);
}