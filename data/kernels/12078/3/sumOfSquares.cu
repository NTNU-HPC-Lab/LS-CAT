#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void sumOfSquares(int* num, int* result, clock_t* time)
{
const int tid = threadIdx.x;
const int size = DATA_SIZE / THREAD_NUM;

int sum = 0;
clock_t start;
if (tid == 0)
start = clock();

for (int i = tid * size; i < (tid + 1) * size; ++i)
{
sum += num[i] * num[i] * num[i];
}

result[tid] = sum;
if (tid == 0)
*time = clock() - start;
}