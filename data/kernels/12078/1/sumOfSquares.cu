#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void sumOfSquares(int* num, int* result, clock_t* time)
{
//shared memory
extern __shared__ int shared[];
const int tid = threadIdx.x;
const int bid = blockIdx.x;
shared[tid] = 0;

if (tid == 0)
time[bid] = clock();

for (int i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM)
{
shared[tid] += num[i] * num[i] * num[i];
}

//synchronized
__syncthreads();

//sum
if (tid == 0)
{
for (int i = 1; i < THREAD_NUM; ++i)
shared[0] += shared[i];

result[bid] = shared[0];
}

//result[bid * THREAD_NUM + tid] = sum;
if (tid == 0)
time[bid + BLOCK_NUM] = clock();
}