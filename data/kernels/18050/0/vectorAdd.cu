#include "hip/hip_runtime.h"
#include "includes.h"
/*-----This is a vector addition--*/
/*---- @ Cuda/c ------*/
/*---- __NS__Bologna__2020__*/




__global__ void vectorAdd(int* a, int* b, int* c, int n){
// calculate index thread
int tid = blockIdx.x * blockDim.x + threadIdx.x;
// Make sure we stay in-bounds
if(tid < n)
// Vector add
c[tid] = a[tid] + b[tid];
}