#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matriMult(int* m, int* n, int* p, int size){
// Calculate Row and Coulmn
int row = blockIdx.y * blockDim.y + threadIdx.y;
int column = blockIdx.x * blockDim.x + threadIdx.x;
int p_sum = 0;
for(int i = 0; i < size; i++){
p_sum += m[row * size + i] * n[i * size + column];
}

p[row * size + column] = p_sum;
}