#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

if ( x < width && y < height ) {
out[y * rowSizeOut + x] = in[y * rowSizeIn + x];
}
}