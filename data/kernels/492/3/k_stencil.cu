#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

// Computes the 1-stencil using GPUs.
// We don't check for error here for brevity.
// In your implementation - you must do it!

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);
__global__ void k_stencil (int *A, int *B, int sizeOfA)
{
extern __shared__ int s[];
// Id of thread in the block.
int localId = threadIdx.x;

// The first index of output element computed by this block.
int startOfBlock = blockIdx.x * blockDim.x * OUTPUT_PER_THREAD;

// The Id of the thread in the scope of the grid.
int globalId = localId + startOfBlock;

if (globalId >= sizeOfA)
return;

// Fetching into shared memory.
for (int i = 0 ; i < OUTPUT_PER_THREAD ; ++i)
{
if (globalId + i*BLOCK_SIZE < sizeOfA)
{
s[localId + i*BLOCK_SIZE] = A[globalId + i*BLOCK_SIZE];
}
}

if (localId < k && blockDim.x*OUTPUT_PER_THREAD + globalId < sizeOfA)
{
s[localId + blockDim.x*OUTPUT_PER_THREAD] =  A[blockDim.x*OUTPUT_PER_THREAD + globalId];
}

// We must sync before reading from shared memory.
__syncthreads();

int sum = 0;
for (int j = 0 ; j < OUTPUT_PER_THREAD ; ++j)
{
sum = 0;
if (globalId + j*BLOCK_SIZE >= sizeOfA - k)
return;
for (int i = 0 ; i < k + 1 ; ++i)
{
sum += s[localId + j*BLOCK_SIZE + i];
}
B[globalId + BLOCK_SIZE*j] = sum ;
}
}