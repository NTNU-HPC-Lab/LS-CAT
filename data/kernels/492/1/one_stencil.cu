#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

// Computes the 1-stencil using GPUs.
// We don't check for error here for brevity.
// In your implementation - you must do it!

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);
__global__ void one_stencil (int *A, int *B, int sizeOfA)
{
extern __shared__ int s[];
// Id of thread in the block.
int localId = threadIdx.x;

// The first index of output element computed by this block.
int startOfBlock = blockIdx.x * blockDim.x;

// The Id of the thread in the scope of the grid.
int globalId = localId + startOfBlock;

if (globalId >= sizeOfA)
return;

// Fetching into shared memory.
s[localId] = A[globalId];
if (localId < 2 && blockDim.x + globalId < sizeOfA)
{
s[blockDim.x + localId] =  A[blockDim.x + globalId];
}

// We must sync before reading from shared memory.
__syncthreads();

// Each thread computes a single output.
if (globalId < sizeOfA - 2)
B[globalId] = s[localId] + s[localId + 1] + s[localId + 2];
}