#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

// Computes the 1-stencil using GPUs.
// We don't check for error here for brevity.
// In your implementation - you must do it!

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);
__global__ void one_stencil_with_rc (int *A, int *B, int sizeOfA)
{
// Declaring local register cache.
int rc[2];

// Id of thread in the warp.
int localId = threadIdx.x % WARP_SIZE;

// The first index of output element computed by this warp.
int startOfWarp = blockIdx.x * blockDim.x + WARP_SIZE*(threadIdx.x / WARP_SIZE);

// The Id of the thread in the scope of the grid.
int globalId = localId + startOfWarp;

if (globalId >= sizeOfA)
return;

// Fetching into shared memory.
rc[0] = A[globalId];
if (localId < 2 && WARP_SIZE + globalId < sizeOfA)
{
rc[1] =  A[WARP_SIZE + globalId];
}

// Each thread computes a single output.
int ac = 0;
int toShare = rc[0];

bool isLastWarp = sizeOfA - startOfWarp < WARP_SIZE;

// The number of threads in the warp which are inactive.
// Possibly bigger than zero only for the last warp.
int inactiveThreadsInWarp = isLastWarp ? startOfWarp + WARP_SIZE - sizeOfA : 0;


// Accessing register cache.
// We use a precomputed active mask.
// This is because otherwise only a subset of active threads return from
//	the __activemask() call, which will resemble a wrong picture of
//	the currently active threads in the warp.
//	notice that the active mask does not change along the following
//	loop so we claculate it just once.
//	Please refer to the cuda developers guide for futher information.
unsigned mask = //__activemask(); <-- Wrong!
(0xffffffff) >> (inactiveThreadsInWarp);

for (int i = 0 ; i < 3 ; ++i)
{
// Threads decide what value will be published in the following access.
if (localId < i)
toShare = rc[1];

ac += __shfl_sync(mask, toShare, (localId + i) % WARP_SIZE);
}

if (globalId < sizeOfA - 2)
B[globalId] = ac;
}