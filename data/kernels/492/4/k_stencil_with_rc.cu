#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

// Computes the 1-stencil using GPUs.
// We don't check for error here for brevity.
// In your implementation - you must do it!

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);
__global__ void k_stencil_with_rc (int *A, int *B, int sizeOfA)
{
int sizeOfB = sizeOfA - k;

// Declaring local register cache.
int rc[LOCAL_REGISTER_SIZE];

// Id of thread in the warp.
int localId = threadIdx.x % WARP_SIZE;

// The first index of output element computed by this warp.
int startOfWarp = (blockIdx.x * blockDim.x + WARP_SIZE*(threadIdx.x / WARP_SIZE))*OUTPUT_PER_THREAD;

// The Id of the thread in the scope of the grid.
int globalId = localId + startOfWarp;

if (globalId >= sizeOfA)
return;

// Fetching into shared memory.

#pragma unroll
for (int i = 0 ; i < OUTPUT_PER_THREAD ; ++i)
{
if (globalId + WARP_SIZE*i >= sizeOfA)
{
continue;
}
rc[i] = A[(int)(globalId + WARP_SIZE*i)];
}

rc[LOCAL_REGISTER_SIZE - 1] =  A[OUTPUT_PER_THREAD*WARP_SIZE + globalId];
// Each thread computes a single output.

bool warpHasInactiveThreads = sizeOfA - startOfWarp < WARP_SIZE;

// The number of threads in the warp which are inactive.
// Possibly bigger than zero only for the last warp.
int inactiveThreadsInWarp = warpHasInactiveThreads ? startOfWarp + WARP_SIZE - sizeOfA : 0;


// Accessing register cache.
// We use a precomputed active mask.
// This is because otherwise only a subset of active threads return from
//	the __activemask() call, which will resemble a wrong picture of
//	the currently active threads in the warp.
//	notice that the active mask does not change along the following
//	loop so we claculate it just once.
//	Please refer to the cuda developers guide for futher information.
unsigned mask = //__activemask(); <-- Wrong!
(0xffffffff) >> (inactiveThreadsInWarp);
#pragma unroll
for (int j = 0 ; j < OUTPUT_PER_THREAD ; ++j)
{
int toShare = rc[j];
int ac = 0;
#pragma unroll
for (int i = 0 ; i < k + 1 ; ++i)
{
// Threads decide what value will be published in the following access.
ac += __shfl_sync(mask, toShare, (localId + i) & (WARP_SIZE - 1));
toShare += (i==localId)*(rc[j+1] - rc[j]);
}

if (globalId + j*WARP_SIZE >= sizeOfB)
{
continue;
}

B[globalId + j*WARP_SIZE] = ac ;

}
}