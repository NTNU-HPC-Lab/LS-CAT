#include "hip/hip_runtime.h"
#include "includes.h"
/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order accurate in time and eigth in space.

Oleg Ovcharenko
Vladimir Kazei, 2019

oleg.ovcharenko@kaust.edu.sa
vladimir.kazei@kaust.edu.sa
*/

/*
Add this to c_cpp_properties.json if linting isn't working for CUDA libraries
"includePath": [
"/usr/local/cuda-10.0/targets/x86_64-linux/include",
"${workspaceFolder}/**"
],
*/

// Check error codes for CUDA functions
__global__ void kernel_add_wavelet(float *d_u, float *d_wavelet, int it)
{
/*
d_u             :pointer to an array on device where to add source term
d_wavelet       :pointer to an array on device with source signature
it              :time step id
*/
unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int idx = gy * c_nx + gx;

if ((gx == c_isrc) && (gy == c_jsrc))
{
d_u[idx] += d_wavelet[it];
}
}