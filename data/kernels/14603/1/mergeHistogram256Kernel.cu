#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/


////////////////////////////////////////////////////////////////////////////////
// Shortcut shared memory atomic addition functions
////////////////////////////////////////////////////////////////////////////////

#define TAG_MASK 0xFFFFFFFFU
__global__ void mergeHistogram256Kernel( uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount )
{
uint sum = 0;

for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
{
sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
}

__shared__ uint data[MERGE_THREADBLOCK_SIZE];
data[threadIdx.x] = sum;

for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
{
__syncthreads();

if (threadIdx.x < stride)
{
data[threadIdx.x] += data[threadIdx.x + stride];
}
}

if (threadIdx.x == 0)
{
d_Histogram[blockIdx.x] = data[0];
}
}