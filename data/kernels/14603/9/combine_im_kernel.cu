#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void combine_im_kernel(const float *A, const float *B, float *C, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

/*
combines images for a joint histogram computation with the formula:
comb_im = B1*(im1 + im2*(B2-1))/(B1*B2 - 1)

for a joint histogram of 256: B1*B2 must equal 256
--> choose B1=B2=16
*/

float B1 = 16.0;
float B2 = 16.0;

if (i < numElements)
{
C[i] = B1*(A[i] + B[i] * (B2 - 1)) / (B1*B2 - 1);
}
}