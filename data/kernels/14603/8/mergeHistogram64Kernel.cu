#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeHistogram64Kernel( float *d_Histogram, float *d_PartialHistograms, uint histogramCount )
{
__shared__ uint data[MERGE_THREADBLOCK_SIZE];

uint sum = 0;

for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
{
sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
}

data[threadIdx.x] = sum;

for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
{
__syncthreads();

if (threadIdx.x < stride)
{
data[threadIdx.x] += data[threadIdx.x + stride];
}
}

if (threadIdx.x == 0)
{
d_Histogram[blockIdx.x] = data[0];
}
}