#include "hip/hip_runtime.h"
#include "includes.h"
#define Tile_size 2


//Function To handle any errors occurred in the function calls
__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
__shared__ float sA[Tile_size][Tile_size];   // Tile size to store elements in shared memory
__shared__ float sB[Tile_size][Tile_size];

int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
int Col = blockDim.x*blockIdx.x + threadIdx.x;
float Cvalue = 0.0;
sA[threadIdx.y][threadIdx.x] = 0.0;
sB[threadIdx.y][threadIdx.x] = 0.0;

for (int k = 0; k < (((numAColumns - 1)/ Tile_size) + 1); k++)
{
if ( (Row < numARows) && (threadIdx.x + (k*Tile_size)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
{
sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*Tile_size)];
}
else
{
sA[threadIdx.y][threadIdx.x] = 0.0;
}
if ( Col < numBColumns && (threadIdx.y + k*Tile_size) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
{
sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*Tile_size)*numBColumns + Col];
}
else
{
sB[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();

for (int j = 0; j < Tile_size; ++j)//Multiplying Elements present in tile
{
Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
}
}
if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
{
C[Row*numCColumns + Col] = Cvalue;
}
}