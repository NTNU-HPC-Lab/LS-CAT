#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 100
#define GRID_SIZE 100
#define N GRID_SIZE * BLOCK_SIZE


__global__ void VectorAdd (int *A, int *B, int *C) {
int x = threadIdx.x + blockIdx.x * blockDim.x;
C[x] = A[x] + B[x];
}