#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DotProd(int *a, int *b, int *c) {

__shared__ int temp[THREADS_PER_BLOCK];

int x = threadIdx.x + blockDim.x * blockIdx.x;
printf("Block ID :%d:\n", blockIdx.x);
printf("Block Dim :%d:\n", blockDim.x);
printf("Theard ID :%d:\n", threadIdx.x);
temp[threadIdx.x] = a[x] * b[x];
printf("Temp:%d\n", temp[threadIdx.x]);

__syncthreads();

if (threadIdx.x == 0)
{
int i,sum = 0;
for (i = 0; i < THREADS_PER_BLOCK; i++)
{
sum += temp[i];
}
printf("\nSUM[%d]:%d", blockIdx.x, sum);
atomicAdd(c, sum);
}
}