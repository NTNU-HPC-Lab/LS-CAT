#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

extern __shared__ int temp[];

temp[threadIdx.x] = in[gid];

for(int offset=1; offset<n; offset=(offset<<1)){
__syncthreads();
if(threadIdx.x >= offset){
temp[threadIdx.x] += temp[threadIdx.x-offset];
} else if(gid >= offset){
temp[threadIdx.x] += in[gid-offset];
}
__syncthreads(); //can only control threads in a block.
in[gid] = temp[threadIdx.x];
}
out[gid] = in[gid];
//out = in;
}