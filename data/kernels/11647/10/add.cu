#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int offset, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

out[gid] = in[gid];
if(gid >= offset)
out[gid] += in[gid-offset];
}