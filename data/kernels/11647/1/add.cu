#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int offset, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

extern __shared__ int temp[];

temp[threadIdx.x] = in[gid];

__syncthreads(); //can only control threads in a block.
if(threadIdx.x >= offset){
in[threadIdx.x] += temp[threadIdx.x-offset];
} else if(gid >= offset){
in[threadIdx.x] += in[gid-offset];
}
in[gid] = temp[threadIdx.x];
}