#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_updatesum(int *array, int *update_array, int size)
{
extern __shared__ int shared[];

unsigned int tid = threadIdx.x;
unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
int op = 0;

if (blockIdx.x > 0) {
op = update_array[blockIdx.x - 1];
}

shared[tid] = array[id] + op;
array[id] = shared[tid];
}