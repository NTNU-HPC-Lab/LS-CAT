#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(double* out, int n){
int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;
out[gid] = 0.0;
}