#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply_cuda(int* d_a, int* d_b, int* d_c, int m, int n) {

int i = blockIdx.y * blockDim.y + threadIdx.y;    // Row i of matrix C
int j = blockIdx.x * blockDim.x + threadIdx.x;    // Column j of matrix C

//Compute c[i][j] = a[i][k]+b[k][j] over k = 0...n-1
int cell = 0;
for (int k=0; k<n; k++)
cell += d_a[i*n+k] * d_b[k*m+j];
d_c[i*m+j]=cell;
}