#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int d, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

int pre = (d==0) ? 1 : (2<<(d-1));

if(gid >= pre) {
in[gid] += in[gid-pre];
}
}