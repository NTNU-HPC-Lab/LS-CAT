#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000 //input data size: 10,000,000
#define BLOCKSIZE 1024

/* prefix sum */

using namespace std;

__global__ void add(double* in, double* out, int offset, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

out[gid] = in[gid];
if(gid >= offset)
out[gid] += in[gid-offset];
}