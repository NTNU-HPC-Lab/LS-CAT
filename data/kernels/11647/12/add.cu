#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

extern __shared__ int temp[];

int pout = 0, pin = 1;
temp[threadIdx.x + pout * n] = (threadIdx.x>0) ? in[threadIdx.x-1] : 0;
__syncthreads();

for(int offset=1; offset<n; offset=(offset<<1)){
int t = pout;
pout = pin;
pin = t;

if(threadIdx.x >= offset){
temp[threadIdx.x + pout*n] += temp[threadIdx.x + pin*n - offset];
} else {
temp[threadIdx.x+pout*n] = temp[threadIdx.x+pin*n];
}
__syncthreads();
}
out[threadIdx.x] = temp[threadIdx.x+pout*n];
}