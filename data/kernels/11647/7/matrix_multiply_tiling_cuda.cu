#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply_tiling_cuda(int* A, int* B, int* C, int m, int n) {
// Block index
int bx = blockIdx.x;
int by = blockIdx.y;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Index of the first sub-matrix of A processed by the block
int aBegin = n * blockDim.y * by;

// Index of the last sub-matrix of A processed by the block
int aEnd   = aBegin + n - 1;

// Step size used to iterate through the sub-matrices of A
int aStep  = blockDim.x;

// Index of the first sub-matrix of B processed by the block
int bBegin = blockDim.x * bx;

// Step size used to iterate through the sub-matrices of B
int bStep  = blockDim.y * m;

// Csub is used to store the element of the block sub-matrix
// that is computed by the thread
int Csub = 0;

// Loop over all the sub-matrices of A and B
// required to compute the block sub-matrix
for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
{

// Declaration of the shared memory array As used to
// store the sub-matrix of A
// Suppose to be As[blockDim.y][blockDim.x] but need dynamic allocation
// For simplicity, use a macro here
__shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

// Declaration of the shared memory array Bs used to
// store the sub-matrix of B
// Suppose to be Bs[blockDim.x][blockDim.y] but need dynamic allocation
// For simplicity, use a macro here
__shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

// Load the matrices from device memory
// to shared memory; each thread loads
// one element of each matrix
As[ty][tx] = A[a + n * ty + tx];
Bs[ty][tx] = B[b + m * ty + tx];

// Synchronize to make sure the matrices are loaded
__syncthreads();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll

for (int k = 0; k < blockDim.x; ++k)
{
Csub += As[ty][k] * Bs[k][tx];
}

// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write the block sub-matrix to device memory;
// each thread writes one element
int c = m * blockDim.y * by + blockDim.x * bx;
C[c + m * ty + tx] = Csub;
}