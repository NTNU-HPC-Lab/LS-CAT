#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(double* in, double* out, int offset, int n){

int gid = threadIdx.x + blockIdx.x * blockDim.x;
if(gid >= n) return ;

out[gid] = in[gid];
if(gid >= offset)
out[gid] += in[gid-offset];
}