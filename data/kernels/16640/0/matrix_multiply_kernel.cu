#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn){
double out;
long i, j;
i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<dim_mn){
out = 0.;
for (j=0; j<dim_mn; j++){
out += matrix[i*dim_mn+j] * vector_in[j];
}
vector_out[i] = out;
}
}