#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;


__global__ void prescan(float *g_odata, float *g_idata, int n)
{
extern __shared__ float temp[];  // allocated on invocation
int thid = threadIdx.x;
int offset = 1;

temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
temp[2 * thid + 1] = g_idata[2 * thid + 1];
//printf("%d - %f - %f \n", thid, g_odata[2 * thid], g_odata[2 * thid + 1]);
//printf("%d - %f - %f \n", thid, g_idata[2 * thid], g_idata[2 * thid + 1]);
for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree
{
__syncthreads();
if (thid < d)
{

int ai = offset*(2 * thid + 1) - 1;
int bi = offset*(2 * thid + 2) - 1;


temp[bi] += temp[ai];
}
offset *= 2;
}


if (thid == 0) { temp[n - 1] = 0; } // clear the last element


for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
{
offset >>= 1;
__syncthreads();
if (thid < d)
{


int ai = offset*(2 * thid + 1) - 1;
int bi = offset*(2 * thid + 2) - 1;


float t = temp[ai];
temp[ai] = temp[bi];
temp[bi] += t;
}
}
__syncthreads();

g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
g_odata[2 * thid + 1] = temp[2 * thid + 1];

//	printf("%d - %f - %f \n", thid, g_odata[2 * thid], g_odata[2 * thid + 1]);
//printf("%d - %f - %f \n", thid, g_idata[2 * thid], g_idata[2 * thid + 1]);
}