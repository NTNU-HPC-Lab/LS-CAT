#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
//@@ Insert code to implement vector addition here
int i = threadIdx.x+blockDim.x*blockIdx.x;
if(i<len) out[i] = in1[i] + in2[i];

}