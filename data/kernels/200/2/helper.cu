#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void helper(float * output, float * blocksum, int len) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < len){
for (int j=0; j<i/blockDim.x; j++)
output[i] += blocksum[j];

}

}