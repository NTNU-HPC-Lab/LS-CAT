#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_image_array_GPU(unsigned long long int* image, int pixels_per_image)
{
int my_pixel = threadIdx.x + blockIdx.x*blockDim.x;
if (my_pixel < pixels_per_image)
{
// -- Set the current pixel to 0 and return, avoiding overflow when more threads than pixels are used:
image[my_pixel] = (unsigned long long int)(0);    // Initialize non-scatter image
my_pixel += pixels_per_image;                     //  (advance to next image)
image[my_pixel] = (unsigned long long int)(0);    // Initialize Compton image
my_pixel += pixels_per_image;                     //  (advance to next image)
image[my_pixel] = (unsigned long long int)(0);    // Initialize Rayleigh image
my_pixel += pixels_per_image;                     //  (advance to next image)
image[my_pixel] = (unsigned long long int)(0);    // Initialize multi-scatter image
}
}