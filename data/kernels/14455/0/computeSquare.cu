#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void computeSquare(int *d_in, int *d_out) {
int index = threadIdx.x;
d_out[index] = d_in[index] * d_in[index];
}