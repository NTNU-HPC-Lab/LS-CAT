#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_max(int* input, int* result, int n)
{
__shared__ int sdata[size];
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int tx = threadIdx.x;
int x = -INT_MAX;

if (i<n)
{
x = input[i];
}
sdata[tx] = x;
__syncthreads();
for(unsigned int s = blockDim.x >> 1 ; s>0 ; s>>=1)
{
__syncthreads();
if(tx<s)
{
if(sdata[tx]>sdata[tx+s])
sdata[tx]=sdata[tx+s];
}
}
if (threadIdx.x == 0)
{
result[blockIdx.x] = sdata[0];
}
}