#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE (100 * 1024 * 1024)



__global__ void histo_kernel_optimization(unsigned char *buffer, int size, unsigned int *histo)
{
__shared__ unsigned int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x + blockDim.x * blockIdx.x;
int stride = blockDim.x * gridDim.x;

while (i < size)
{
atomicAdd(&histo[buffer[i]], 1);
i += stride;
}
__syncthreads();
atomicAdd(&histo[threadIdx.x], temp[threadIdx.x]);
}