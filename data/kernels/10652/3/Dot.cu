#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Dot(float *a, float *b, float *c)
{
__shared__ float cache[ThreadsPerBlock];
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int cacheIndex = threadIdx.x;
float temp = 0;
while (tid < N) {
temp += a[tid] * b[tid];
tid += blockDim.x * gridDim.x;
}
cache[cacheIndex] = temp;
__syncthreads();

int i = blockDim.x / 2;
while (i != 0)
{
if (cacheIndex < i)
cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
i /= 2;
}

if (cacheIndex == 0)
c[blockIdx.x] = cache[0];
}