#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE (100 * 1024 * 1024)



__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
int stride = blockDim.x * gridDim.x;

while (i < size)
{
atomicAdd(&histo[buffer[i]], 1);
i += stride;
}
}