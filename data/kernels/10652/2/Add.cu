#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(int *a, int *b, int *c)
{
int tid = threadIdx.x;

if (tid < N)
c[tid] = a[tid] + b[tid];
}