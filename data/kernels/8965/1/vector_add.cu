#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_add(int *a, int *b, int *c){
int index =  blockIdx.x * blockDim.x + threadIdx.x;
c[index] = a[index] + b[index] ;
}