#include "hip/hip_runtime.h"
#include "includes.h"
/***
__shared__  通过共享内存来完成线程间的通信
这一段代码 通过共享内存
***/


hipError_t addWithCuda(int *c, const int *a, size_t size);


__global__ void addKernel(int *c, const int *a){
int i = threadIdx.x;
extern __shared__ int seme []; //声明一个全局的 共享内存的变量
seme[i] = a[i];
__syncthreads();  //同一个块的线程同步  等待seme将所有数据加载进来
if(i==0){ //第一个线程进行二次方
c[0] = 0;
for (int d=0; d<5; d++){
printf("seme[d] * seme [d] %d \n", d);
c[0] += seme[d] * seme [d];
}
printf("给 seme 赋值 %d ", i);
seme[i] = 0;
}
if(i==1){
c[1] = 0;
for (int d=0; d<5; d++){
printf("c[1] += seme[d] %d \n", d);
c[1] += seme[d];
}
printf("给 seme 赋值 %d ", i);
seme[i] = 0;
}
if(i==2){
c[2] = 1;
for(int d=0; d<5; d++){
printf("c[2] *= seme[d] %d \n", d);
c[2] *= seme[d];
}
printf("给 seme 赋值 %d ", i);
seme[i] = 0;
}
}