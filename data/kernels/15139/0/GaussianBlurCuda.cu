#include "hip/hip_runtime.h"
#include "includes.h"
//iojpegparts.cu


__global__ void GaussianBlurCuda (unsigned char *pic, unsigned char * outpic, double *mask, int *size){ // size: width, height, mask_width
int pxPosCen = blockIdx.x * blockDim.x + threadIdx.x;
if (pxPosCen >= size[0]*size[1] || pxPosCen < 0) return;
int row, col, x, y, pos;
row = pxPosCen/size[0]; // pixel position taken as width major
col = pxPosCen%size[0];
double sumout[3];
sumout[0] = 0;
sumout[1] = 0;
sumout[2] = 0;
if (row < size[2]/2 || row >= (size[1] - (size[2]/2))) return;
if (col < size[2]/2 || col >= (size[0] - (size[2]/2))) return;
for (int i=0;i<size[2]*size[2];i++){
x = i%size[2] + col - size[2]/2;
y = i/size[2] + row - size[2]/2;
pos = (y*size[0]  + x)*3;
sumout[0]+=(double)(*(pic+pos  )) * mask[i];
sumout[1]+=(double)(*(pic+pos+1)) * mask[i];
sumout[2]+=(double)(*(pic+pos+2)) * mask[i];
}
pos = pxPosCen*3;
*(outpic+pos) = (unsigned char) sumout[0];
*(outpic+pos+1) = (unsigned char) sumout[1];
*(outpic+pos+2) = (unsigned char) sumout[2];
}