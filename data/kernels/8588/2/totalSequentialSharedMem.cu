#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalSequentialSharedMem(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
int tid = threadIdx.x, i = blockIdx.x * blockDim.x;
__shared__ float sdata[BLOCK_SIZE];
sdata[tid] = i + tid < len ? input[i+tid] : 0.0;

if(tid == 0) {
for(unsigned int j = 1; j <blockDim.x; j++)
{
sdata[0] += sdata[j];
}
output[blockIdx.x] = sdata[0];
}
}