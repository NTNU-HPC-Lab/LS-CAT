#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSyncInterleaved(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
int tid = threadIdx.x, i = blockIdx.x * blockDim.x + threadIdx.x;

for(unsigned int j = 1; j <blockDim.x; j *= 2)
{
if (tid % (2 * j) == 0)
input[i] += input[i+j];
__syncthreads();
}

if(tid == 0)
{
output[blockIdx.x] = input[i];
}
}