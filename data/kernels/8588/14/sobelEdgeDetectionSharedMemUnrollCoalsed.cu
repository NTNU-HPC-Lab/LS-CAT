#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMemUnrollCoalsed(int *input, int *output, int width, int height, int thresh) {

__shared__ int shMem[4 * _TILESIZE_2 * _TILESIZE_2 ];

int num = _UNROLL_;
int size = num * _TILESIZE_2;

int i = blockIdx.x * (num * _TILESIZE_) + threadIdx.x;
int j = blockIdx.y * (num * _TILESIZE_) + threadIdx.y;

int xind = threadIdx.x;
int yind = threadIdx.y;

for(int x = 0; x < num; x++)
{
for(int y = 0; y < num; y++)
{
int xOffset = x * (_TILESIZE_), yOffset = y * (_TILESIZE_);
shMem[ size * (yind + yOffset) + (xind + xOffset)] = input[(j + yOffset) * width + (i + xOffset)];
}
}

__syncthreads();

if (i < width - _TILESIZE_ && j < height - _TILESIZE_ && xind > 0 && yind > 0 && xind < (_TILESIZE_2 - 1) && yind < (_TILESIZE_2 - 1))
{
for(int x = 0; x < num; x++)
{
for(int y = 0; y < num; y++)
{
int xOffset = x * _TILESIZE_, yOffset = y * _TILESIZE_;

int sum1 = shMem[(xind + 1 + xOffset) + size * (yind - 1 + yOffset)] -     shMem[(xind - 1 + xOffset) + size * (yind - 1 + yOffset)]
+ 2 * shMem[(xind + 1 + xOffset) + size * (yind     + yOffset)] - 2 * shMem[(xind - 1 + xOffset) + size * (yind     + yOffset)]
+     shMem[(xind + 1 + xOffset) + size * (yind + 1 + yOffset)] -     shMem[(xind - 1 + xOffset) + size * (yind + 1 + yOffset)];

int sum2 = shMem[(xind - 1 + xOffset) + size * (yind - 1 + yOffset)] + 2 * shMem[(xind     + xOffset) + size * (yind - 1 + yOffset)] + shMem[(xind + 1 + xOffset) + size * (yind - 1 + yOffset)]
- shMem[(xind - 1 + xOffset) + size * (yind + 1 + yOffset)] - 2 * shMem[(xind     + xOffset) + size * (yind + 1 + yOffset)] - shMem[(xind + 1 + xOffset) + size * (yind + 1 + yOffset)];

int magnitude = sum1 * sum1 + sum2 * sum2;

int index = (j + yOffset) * width + (i + xOffset);

if(magnitude > thresh)
output[index] = 255;
else
output[index] = 0;

}
}
}

}