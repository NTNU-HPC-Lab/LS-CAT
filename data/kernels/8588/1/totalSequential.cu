#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalSequential(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
int tid = threadIdx.x, i = blockIdx.x * blockDim.x;

if(tid == 0) {
int sum = 0;
for(unsigned int j = 0; j <blockDim.x; j++)
{
sum += input[i + j];
}
output[blockIdx.x] = sum;
}
}