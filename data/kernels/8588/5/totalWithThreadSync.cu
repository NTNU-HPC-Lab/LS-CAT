#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSync(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
int tid = threadIdx.x, i = blockIdx.x * blockDim.x + threadIdx.x;

for(unsigned int j = blockDim.x/2; j > 0; j = j/2)
{
if(tid < j)
{
if ((i + j) < len)
input[i] += input[i+j];
else
input [i] += 0.0;
}
__syncthreads();
}

if(tid == 0)
{
output[blockIdx.x] = input[i];
}
}