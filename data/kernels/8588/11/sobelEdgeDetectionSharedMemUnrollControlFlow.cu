#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMemUnrollControlFlow(int *input, int *output, int width, int height, int thresh) {

unsigned int blockSize = 32;
static __shared__ int shMem[34][34];

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

int xind = threadIdx.x + 1;
int yind = threadIdx.y + 1;

shMem[xind][yind] = input[width * j + i];

if ( i > 0 && j > 0 && i < width - 1 && j < height - 1)
{
if(threadIdx.x == 0)
shMem[xind-1][yind] = input[width * j + i-1];

if(threadIdx.y == 0)
shMem[xind][yind-1] = input[width * (j-1) + i];

if(threadIdx.x == blockSize+1)
shMem[xind+1][yind] = input[width * j + i+1];

if(threadIdx.y == blockSize+1)
shMem[xind][yind+1] = input[width * (j+1) + i];

if(threadIdx.x == 0 && threadIdx.y == 0)
shMem[xind-1][yind-1] = input[width * (j-1) + i-1];

if(threadIdx.x == blockSize+1 && threadIdx.y == 0)
shMem[xind+1][yind-1] = input[width * (j-1) + i+1];

if(threadIdx.x == 0 && threadIdx.y == blockSize+1)
shMem[xind-1][yind+1] = input[width * (j+1) + i-1];

if(threadIdx.x == blockSize+1 && threadIdx.y == blockSize+1)
shMem[xind+1][yind+1] = input[width * (j+1) + i+1];
}
__syncthreads();


int sum1 = 0, sum2 = 0, magnitude;
int num = 3;

for(int xind = 1; xind < num; xind++)
{
for(int yind = 1; yind < num; yind++)
{
sum1 = shMem[xind+1][yind-1] -     shMem[xind-1][yind-1]
+ 2 * shMem[xind+1][yind  ] - 2 * shMem[xind-1][yind  ]
+     shMem[xind+1][yind+1] -     shMem[xind-1][yind+1];

sum2 = shMem[xind-1][yind-1] + 2 * shMem[xind][yind-1] + shMem[xind+1][yind-1]
- shMem[xind-1][yind+1] - 2 * shMem[xind][yind+1] - shMem[xind+1][yind+1];

magnitude = sum1 * sum1 + sum2 * sum2;

if(magnitude > thresh)
output[(j + yind - 1) * width + (i + xind - 1)] = 255;
else
output[(j + yind - 1) * width + (i + xind - 1)] = 0;

}
}

}