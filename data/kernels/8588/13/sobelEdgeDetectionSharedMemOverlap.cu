#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMemOverlap(int *input, int *output, int width, int height, int thresh) {

static __shared__ int shMem[_TILESIZE_2 * _TILESIZE_2];

int blocksize = _TILESIZE_2;
int i = blockIdx.x * (_TILESIZE_) + threadIdx.x;
int j = blockIdx.y * (_TILESIZE_) + threadIdx.y;
int index = j * width + i;

int xind = threadIdx.x;
int yind = threadIdx.y;

shMem[blocksize * yind + xind] = input[index];
__syncthreads();

if ( xind > 0 && yind > 0 && xind < (blocksize - 1) && yind < (blocksize - 1))
{

int sum1 = shMem[xind + 1 + blocksize * (yind - 1)] -     shMem[xind - 1 + blocksize * (yind - 1)]
+ 2 * shMem[xind + 1 + blocksize * (yind    )] - 2 * shMem[xind - 1 + blocksize * (yind    )]
+     shMem[xind + 1 + blocksize * (yind + 1)] -     shMem[xind - 1 + blocksize * (yind + 1)];

int sum2 = shMem[xind - 1 + blocksize * (yind - 1)] + 2 * shMem[xind     + blocksize * (yind - 1)] + shMem[xind + 1 + blocksize * (yind - 1)]
- shMem[xind - 1 + blocksize * (yind + 1)] - 2 * shMem[xind     + blocksize * (yind + 1)] - shMem[xind + 1 + blocksize * (yind + 1)];

int magnitude = sum1 * sum1 + sum2 * sum2;
if(magnitude > thresh)
output[index] = 255;
else
output[index] = 0;
}
}