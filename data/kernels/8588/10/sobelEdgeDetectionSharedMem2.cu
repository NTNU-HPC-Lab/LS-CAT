#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMem2(int *input, int *output, int width, int height, int thresh) {

int regArr[4][4];

int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
int j = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

if ( i > 0 && j > 0 && i < width - 1 && j < height - 1)
{

regArr[0][0] = input[width * (j-1) + i - 1];
regArr[0][1] = input[width * (j-1) + i    ];
regArr[0][2] = input[width * (j-1) + i + 1];
regArr[0][3] = input[width * (j-1) + i + 2];
regArr[1][0] = input[width * (j)   + i - 1];
regArr[1][1] = input[width * (j)   + i    ];
regArr[1][2] = input[width * (j)   + i + 1];
regArr[1][3] = input[width * (j)   + i + 2];
regArr[2][0] = input[width * (j+1) + i - 1];
regArr[2][1] = input[width * (j+1) + i    ];
regArr[2][2] = input[width * (j+1) + i + 1];
regArr[2][3] = input[width * (j+1) + i + 2];
regArr[3][0] = input[width * (j+2) + i - 1];
regArr[3][1] = input[width * (j+2) + i    ];
regArr[3][2] = input[width * (j+2) + i + 1];
regArr[3][3] = input[width * (j+2) + i + 2];

__syncthreads();


int sum1 = 0, sum2 = 0, magnitude;
int num = 3;

for(int xind = 1; xind < num; xind++)
{
for(int yind = 1; yind < num; yind++)
{
sum1 = regArr[xind+1][yind-1] -     regArr[xind-1][yind-1]
+ 2 * regArr[xind+1][yind  ] - 2 * regArr[xind-1][yind  ]
+     regArr[xind+1][yind+1] -     regArr[xind-1][yind+1];

sum2 = regArr[xind-1][yind-1] + 2 * regArr[xind][yind-1] + regArr[xind+1][yind-1]
- regArr[xind-1][yind+1] - 2 * regArr[xind][yind+1] - regArr[xind+1][yind+1];

magnitude = sum1 * sum1 + sum2 * sum2;

if(magnitude > thresh)
output[(j + yind - 1) * width + (i + xind - 1)] = 255;
else
output[(j + yind - 1) * width + (i + xind - 1)] = 0;

}
}
}
}