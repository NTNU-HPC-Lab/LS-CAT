#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSyncAndSharedMem(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
__shared__ float sdata[BLOCK_SIZE];
int tid = threadIdx.x, i = blockIdx.x * blockDim.x + threadIdx.x;

if(tid < len)
sdata[tid] = input[i];
else
sdata[tid] = 0.0;

__syncthreads();

for(unsigned int j = blockDim.x/2; j > 0; j = j/2)
{
if(tid < j)
{
sdata[tid] += sdata[tid+j];
}
__syncthreads();
}

if(tid == 0)
{
output[blockIdx.x] = sdata[0];
}
}