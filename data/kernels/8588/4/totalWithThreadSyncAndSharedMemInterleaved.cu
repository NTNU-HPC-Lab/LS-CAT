#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSyncAndSharedMemInterleaved(float *input, float *output, int len) {
//@@ Compute reduction for a segment of the input vector
__shared__ float sdata[BLOCK_SIZE];
int tid = threadIdx.x, i = blockIdx.x * blockDim.x + threadIdx.x;

if(i  < len)
sdata[tid] = input[i];
else
sdata[tid] = 0.0;

for(unsigned int j = 1; j < blockDim.x; j *= 2)
{
if (tid % (2 * j) == 0)
sdata[tid] += sdata[tid+j];
__syncthreads();
}

if(tid == 0)
{
output[blockIdx.x] = sdata[0];
}
}