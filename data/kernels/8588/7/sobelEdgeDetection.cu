#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetection(int *input, int *output, int width, int height, int thresh) {

int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
int index = j * width + i;

if ( ((i > 0) && (j > 0)) && ((i < (width - 1)) && (j < (height - 1))))
{

int sum1 = 0, sum2 = 0, magnitude;

sum1 = input[width * (j - 1) + (i + 1)] -     input[width * (j - 1) + (i - 1)]
+ 2 * input[width * (j)     + (i + 1)] - 2 * input[width * (j)     + (i - 1)]
+     input[width * (j + 1) + (i + 1)] -     input[width * (j + 1) + (i - 1)];

sum2 = input[width * (j - 1) + (i - 1)] + 2 * input[width * (j - 1) + (i)] + input[width * (j - 1) + (i + 1)]
- input[width * (j + 1) + (i - 1)] - 2 * input[width * (j + 1) + (i)] - input[width * (j + 1) + (i + 1)];

magnitude = sum1 * sum1 + sum2 * sum2;
if(magnitude > thresh)
output[index] = 255;
else
output[index] = 0;
}
}