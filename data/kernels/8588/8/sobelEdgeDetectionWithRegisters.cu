#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionWithRegisters (int *input, int *output, int width, int height, int thresh) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
int index = j * width + i;

int val1 = input[width * (j - 1) + (i + 1)], val2 = input[width * (j - 1) + (i - 1)], val3 = input[width * (j + 1) + (i + 1)], val4 = input[width * (j + 1) + (i - 1)];

if ( ((i > 0) && (j > 0)) && ((i < (width - 1)) && (j < (height - 1))))
{

int sum1 = 0, sum2 = 0, magnitude;

sum1 = val1 - val2
+ 2 * input[width * (j)     + (i + 1)] - 2 * input[width * (j)     + (i - 1)]
+     val3 - val4;

sum2 = val2 + 2 * input[width * (j - 1) + (i)] + val1
- val4 - 2 * input[width * (j + 1) + (i)] - val3;

magnitude = sum1 * sum1 + sum2 * sum2;
if(magnitude > thresh)
output[index] = 255;
else
output[index] = 0;
}
else {
output[index] = 0;
}
}