#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMemUnroll(int *input, int *output, int width, int height, int thresh) {

__shared__ int shMem[4 * _TILESIZE_2 * _TILESIZE_2 ];

int num = _UNROLL_;
int size = num * _TILESIZE_2;

int i = blockIdx.x * num * _TILESIZE_ + threadIdx.x * num;
int j = blockIdx.y * num * _TILESIZE_ + threadIdx.y * num;

int xind = num * threadIdx.x;
int yind = num * threadIdx.y;

for(int x = 0; x < num; x++)
{
for(int y = 0; y < num; y++)
{
shMem[ size * (yind + y) + (xind + x)] = input[(j + y) * width + (i + x)];
}
}

__syncthreads();

if ( xind > 0 && yind > 0 && xind < (size - 2) && yind < (size - 2))
{
for(int x = 0; x < num; x++)
{
for(int y = 0; y < num; y++)
{

int sum1 = shMem[(xind + 1 + x) + size * (yind - 1 + y)] -     shMem[(xind - 1 + x) + size * (yind - 1 + y)]
+ 2 * shMem[(xind + 1 + x) + size * (yind     + y)] - 2 * shMem[(xind - 1 + x) + size * (yind     + y)]
+     shMem[(xind + 1 + x) + size * (yind + 1 + y)] -     shMem[(xind - 1 + x) + size * (yind + 1 + y)];

int sum2 = shMem[(xind - 1 + x) + size * (yind - 1 + y)] + 2 * shMem[(xind     + x) + size * (yind - 1 + y)] + shMem[(xind + 1 + x) + size * (yind - 1 + y)]
- shMem[(xind - 1 + x) + size * (yind + 1 + y)] - 2 * shMem[(xind     + x) + size * (yind + 1 + y)] - shMem[(xind + 1 + x) + size * (yind + 1 + y)];

int magnitude = sum1 * sum1 + sum2 * sum2;

int index = (j + y) * width + (i + x);

if(magnitude > thresh)
output[index] = 255;
else
output[index] = 0;

}
}
}

}