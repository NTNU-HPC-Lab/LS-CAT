#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
for (i = 0; i < n; ++i) {
int val = input[i*stride];
largest = (val>largest) ? val : largest;
}
for (i = 0; i < n; ++i) {
float e = expf(input[i*stride] / temp - largest / temp);
sum += e;
output[i*stride] = e;
}
for (i = 0; i < n; ++i) {
output[i*stride] /= sum;
}
}
__device__ void softmax_device(int n, float *input, float temp, float *output)
{
int i;
float sum = 0;
float largest = -INFINITY;
for(i = 0; i < n; ++i){
int val = input[i];
largest = (val>largest) ? val : largest;
}
for(i = 0; i < n; ++i){
float e = exp(input[i]/temp - largest/temp);
sum += e;
output[i] = e;
}
for(i = 0; i < n; ++i){
output[i] /= sum;
}
}
__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (id >= spatial*batch*groups) return;
int s = id % spatial;
id = id / spatial;
int g = id % groups;
int b = id / groups;
int goff = group_offset[g] * spatial;
int boff = b*stride;
softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}