#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_touch(uint64_t *x, const size_t size)
{
const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size)
{
x[i] = 0;
}
}