#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_activateTanh(double* pA, int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n) {
pA[id] = tanh(pA[id]);
}
}