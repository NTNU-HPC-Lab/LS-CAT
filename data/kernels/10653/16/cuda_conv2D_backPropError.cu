#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_conv2D_backPropError(double* upStreamError, const double* err, const double* pKernels, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputChannels, size_t inputRows, size_t inputCols, size_t padding, size_t stride)
{
// Do all values for i, j, and k in parallel
int id = blockIdx.x * blockDim.x + threadIdx.x;
size_t i = id % inputChannels;
id /= inputChannels;
size_t j = id % outputRows;
id /= outputRows;
if(id >= kernelCount)
return;
size_t k = id;

// Compute some intermediate values
size_t outChannelOffset = k * outputRows * outputCols;
size_t outRowOffset = j * outputCols;
int inRowOffset = j * stride - padding;

// This block of code is derived from the serial implementation
size_t kk = k * inputChannels * kernelRows * kernelCols;
size_t index = outChannelOffset + outRowOffset + i;
int inColOffset = i * stride - padding;
for(size_t z = 0; z < inputChannels; z++)
{
size_t kernelChannelOffset = z * kernelRows * kernelCols;
size_t inChannelOffset = z * inputRows * inputCols;
for(size_t y = 0; y < kernelRows; y++)
{
size_t kernelRowOffset = y * kernelCols;
int inRow = inRowOffset + y;
for(size_t x = 0; x < kernelCols; x++)
{
int inCol = inColOffset + x;
if(inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputRows)
{
size_t idx = inChannelOffset + inputCols * inRow + inCol;
upStreamError[idx] += pKernels[kk + kernelChannelOffset + kernelRowOffset + x] * err[index];
}
}
}
}

}