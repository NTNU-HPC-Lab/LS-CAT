#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_deactivateBend(double* pE, const double* pA, int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n) {
double x = pE[id];
pE[id] *= 0.5 * (x / sqrt(x * x + 1)) + 1;
}
}