#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_fill(double* pVec, double val, int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n)
pVec[n] = val;
}