#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_conv2D_ff(double* pA, double* pNet, const double* in, const double* pKernels, const double* pBias, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputRows, size_t inputCols, size_t inputChannels, size_t padding, size_t stride)
{
// Do all values for i, j, and k in parallel
int id = blockIdx.x * blockDim.x + threadIdx.x;
size_t i = id % outputCols;
id /= outputCols;
size_t j = id % outputRows;
id /= outputRows;
if(id >= kernelCount)
return;
size_t k = id;

// Compute some intermediate values
size_t outChannelOffset = k * outputRows * outputCols;
size_t outRowOffset = j * outputCols;
int inRowOffset = j * stride - padding;

// This block of code is derived from the serial implementation
size_t kk = k * inputChannels * kernelRows * kernelCols;
size_t index = outChannelOffset + outRowOffset + i;
int inColOffset = i * stride - padding;
pNet[index] = pBias[k];
for(size_t z = 0; z < inputChannels; z++)
{
size_t kernelChannelOffset = z * kernelRows * kernelCols;
size_t inChannelOffset = z * inputRows * inputCols;
for(size_t y = 0; y < kernelRows; y++)
{
size_t kernelRowOffset = y * kernelCols;
int inRow = inRowOffset + y;
for(size_t x = 0; x < kernelCols; x++)
{
int inCol = inColOffset + x;
if(inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputRows)
{
size_t idx = inChannelOffset + inputCols * inRow + inCol;
pNet[index] += pKernels[kk + kernelChannelOffset + kernelRowOffset + x] * in[idx];
}
}
}
}

//a[index] = pThis->m_pActivationFunction->squash(n[index]);
pA[index] = tanh(pNet[index]);
}