#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_deactivateTanh(double* pE, const double* pA, int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n) {
pE[id] *= (1.0 - (pA[id] * pA[id]));
}
}