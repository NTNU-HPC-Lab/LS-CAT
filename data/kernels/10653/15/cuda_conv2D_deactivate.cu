#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_conv2D_deactivate(double* err, const double* net, const double* activation, size_t outputs)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id >= outputs)
return;
err[id] *= (1.0 - activation[id] * activation[id]);
}