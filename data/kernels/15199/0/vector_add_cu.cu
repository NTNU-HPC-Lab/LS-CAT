#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void vector_add_cu(float *out, float *a, float *b, int n){
for(int i = 0; i < n; i++){
out[i] = a[i] + b[i];
}
}