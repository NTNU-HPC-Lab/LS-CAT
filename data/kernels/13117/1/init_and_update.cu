#include "hip/hip_runtime.h"
#include "includes.h"
/**********************************************************************
* DESCRIPTION:
*   Serial Concurrent Wave Equation - C Version
*   This program implements the concurrent wave equation
*********************************************************************/

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

#define BLOCK_SIZE 512

void check_param(void);
void printfinal (void);


/**********************************************************************
*     Initialize points on line
*********************************************************************/

/**********************************************************************
*     Update all values along line a specified number of times
*********************************************************************/
__global__ void init_and_update (float *values_d, int tpoints, int nsteps){
int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;

if(idx <= 1 || idx >= tpoints)
return;

float old_v, v, new_v;

float x, tmp;
tmp = tpoints - 1;
x = idx / tmp;

v = sin(2.0 * PI * x);
old_v = v;

for (int i = 1; i <= nsteps; i++){
new_v = (2.0 * v) - old_v + (0.09 * (-2.0 * v));
old_v = v;
v = new_v;
}

values_d[idx] = v;

}