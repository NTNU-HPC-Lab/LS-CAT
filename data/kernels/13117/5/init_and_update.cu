#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_and_update (float *values_d, int tpoints, int nsteps){
int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;

if(idx <= 1 || idx >= tpoints)
return;

float old_v, v, new_v;

float x, tmp;
tmp = tpoints - 1;
x = (float)(idx - 1) / tmp;

v = sin(2.0f * PI * x);
old_v = v;

for (int i = 1; i <= nsteps; i++){
new_v = (2.0f * v) - old_v + (0.09f * (-2.0f * v));
old_v = v;
v = new_v;
}

values_d[idx] = v;

}