#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_interpolate_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, const float* weight, float* output)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<N*C;j+=blockDim.x)
{
int n = j/C;
int c = j%C;
int nnSize = nnCount[i*N+n];
for(int k=0;k<nnSize;k++)
{
int m = nnIndex[i*N*K+n*K+k];
float w = weight[i*N*K+n*K+k];
output[i*N*C+j] += input[i*M*C+m*C+c]*w;
}
}
}
}