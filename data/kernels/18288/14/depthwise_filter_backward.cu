#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void depthwise_filter_backward(int B, int N, int M, int F, int C, int r, int K, const int* nnIndex, const int* nnCount, const int* binIndex, const float* input, const float* gradOutput, float* gradFilter, int sharedMemSize, int startIdx)
{
extern __shared__ float gradPerBlock[]; // the gradient on each block
for (int i=threadIdx.x;i<sharedMemSize;i+=blockDim.x)
{
gradPerBlock[i] = 0; // for 1D block
}
__syncthreads();

int endIdx = sharedMemSize+startIdx;
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=blockIdx.y*blockDim.x+threadIdx.x;j<M*(C*r);j+=blockDim.x*gridDim.y)
{
int cout = j%(C*r); // output channel ID
int cin = cout/r;   // input channel ID
int m = j/(C*r);    // output point ID
int nnSize = nnCount[i*M+m];

for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k];   // input point ID
int f = binIndex[i*M*K+m*K+k];

float derFilt = gradOutput[i*M*C*r+j]*input[i*N*C+n*C+cin]/nnSize;

int currIdx = f*C*r+cout;
if((currIdx>=startIdx) && (currIdx<endIdx)) // within the shared memory
{
atomicAdd(&gradPerBlock[currIdx-startIdx],derFilt);
}
}
}
}
__syncthreads();

for (int i=threadIdx.x;i<sharedMemSize;i+=blockDim.x)
{
atomicAdd(&gradFilter[i+startIdx],gradPerBlock[i]); // for 1D block
}
}