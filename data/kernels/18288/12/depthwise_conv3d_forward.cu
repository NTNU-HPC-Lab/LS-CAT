#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void depthwise_conv3d_forward(int B, int N, int M, int C, int r, int K, const int* nnIndex, const int* nnCount, const int* binIndex, const float* input, const float* filter, float* output)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=blockIdx.y*blockDim.x+threadIdx.x;j<M*(C*r);j+=blockDim.x*gridDim.y)
{
int cout = j%(C*r); // output channel ID
int cin = cout/r;   // input channel ID
int m = j/(C*r);    // output point ID
int nnSize = nnCount[i*M+m];

for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k];   // input point ID
int f = binIndex[i*M*K+m*K+k];

output[i*M*C*r+j] += input[i*N*C+n*C+cin]*filter[f*C*r+cout]/nnSize;
}
}
}
}