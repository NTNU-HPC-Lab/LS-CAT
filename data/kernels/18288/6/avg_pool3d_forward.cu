#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void avg_pool3d_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, float* output)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<M*C;j+=blockDim.x)
{
int m = j/C;
int c = j%C;
int nnSize = nnCount[i*M+m];
for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k];
output[i*M*C+j] += input[i*N*C+n*C+c]/nnSize;
}
}
}
}