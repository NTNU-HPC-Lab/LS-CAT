#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void avg_pool3d_backward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* gradOutput, float* gradInput)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<M*C;j+=blockDim.x)
{
int m = j/C;
int c = j%C;
int nnSize = nnCount[i*M+m];
for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k]; // only neighbor, no bin indices, dimension=(B,M,K)
atomicAdd(&gradInput[i*N*C+n*C+c],gradOutput[i*M*C+j]/nnSize);
}
}
}
}