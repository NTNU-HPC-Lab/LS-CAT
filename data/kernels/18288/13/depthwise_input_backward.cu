#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void depthwise_input_backward(int B, int N, int M, int F, int C, int r, int K, const int* nnIndex, const int* nnCount, const int* binIndex, const float* input, const float* filter, const float* gradOutput, float* gradInput)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=blockIdx.y*blockDim.x+threadIdx.x;j<M*(C*r);j+=blockDim.x*gridDim.y)
{
int cout = j%(C*r); // output channel ID
int cin = cout/r;   // input channel ID
int m = j/(C*r);    // output point ID
int nnSize = nnCount[i*M+m];

for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k];   // input point ID
int f = binIndex[i*M*K+m*K+k];

float derIn = gradOutput[i*M*C*r+j]*filter[f*C*r+cout]/nnSize;
atomicAdd(&gradInput[i*N*C+n*C+cin],derIn);
}
}
}
}