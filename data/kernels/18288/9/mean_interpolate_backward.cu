#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_interpolate_backward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* gradOutput, float* gradInput)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<N*C;j+=blockDim.x)
{
int n = j/C;
int c = j%C;
int nnSize = nnCount[i*N+n];
for(int k=0;k<nnSize;k++)
{
int m = nnIndex[i*N*K+n*K+k];
atomicAdd(&gradInput[i*M*C+m*C+c],gradOutput[i*N*C+j]/nnSize);
}
}
}
}