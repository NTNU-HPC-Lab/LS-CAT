#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_pool3d_backward(int B, int N, int M, int C, const int* maxIndex, const float* gradOutput, float* gradInput)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<M*C;j+=blockDim.x)
{
int c = j%C;
int n = maxIndex[i*M*C+j];
atomicAdd(&gradInput[i*N*C+n*C+c],gradOutput[i*M*C+j]);
}
}
}