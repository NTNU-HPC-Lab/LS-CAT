#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_pool3d_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, float* output, int* maxIndex)
{
for(int i=blockIdx.x;i<B;i+=gridDim.x)
{
for(int j=threadIdx.x;j<M*C;j+=blockDim.x)
{
int m = j/C;
int c = j%C;
int nnSize = nnCount[i*M+m];

for(int k=0;k<nnSize;k++)
{
int n = nnIndex[i*M*K+m*K+k];
if (k==0)
{
output[i*M*C+j] = input[i*N*C+n*C+c];
maxIndex[i*M*C+j] = n;
continue;
}

if (input[i*N*C+n*C+c]>output[i*M*C+j])
{
output[i*M*C+j] = input[i*N*C+n*C+c];
maxIndex[i*M*C+j] = n;
}
}
}
}
}