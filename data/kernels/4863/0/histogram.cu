#include "hip/hip_runtime.h"
#include "includes.h"

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
__device__ uint bfe(uint x, uint start, uint nbits)
{
uint bits;
asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
return bits;
}
__global__ void histogram(int* Hist, long long* arrayofkeys, long long size,int noofpartitions)
{
register int thd = threadIdx.x;
register int bD = blockDim.x;
register int bI = blockIdx.x;
uint h,start,nbits;

long long thdindex= bD * bI + thd;
extern __shared__ int sharedpartitions[];
int * sharedHist = (int *)&sharedpartitions[noofpartitions];

for(int m =thd;m<noofpartitions;m=m+bD)
sharedHist[m]=0;

__syncthreads();

start=0;
nbits=(uint)ceil(log2((float)noofpartitions));
if(thdindex<size)
{

h=bfe(arrayofkeys[thdindex],start,nbits);
atomicAdd(&(sharedHist[h]),1);
}
__syncthreads();

for(int n=thd;n<noofpartitions;n=n+bD)
atomicAdd(&(Hist[n]),(sharedHist[n]));

}