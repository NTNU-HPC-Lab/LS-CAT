#include "hip/hip_runtime.h"
#include "includes.h"

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
__global__ void prefixScan(int* Hist,int* Hist_dev_pre, int noofpartitions,long long size)
{
extern __shared__ int sharedpartitions[];
register int thd = threadIdx.x;
int offset = 1;

sharedpartitions[2*thd]=Hist[2*thd];
sharedpartitions[2*thd+1]=Hist[2*thd + 1];

for(int i = noofpartitions>>1;i>0;i>>=1)
{
__syncthreads();
if(thd<i)
{
int x = offset*(2*thd+1)-1;
int y = offset*(2*thd+2)-1;

sharedpartitions[y]+=sharedpartitions[x];
}
offset*=2;
}

if(thd==0){sharedpartitions[noofpartitions-1]=0;}

for(int i = 1;i<noofpartitions;i*=2)
{
offset>>=1;
__syncthreads();
if(thd<i)
{

int x = offset*(2*thd+1)-1;
int y = offset*(2*thd+2)-1;

int tmp = sharedpartitions[x];
sharedpartitions[x]=sharedpartitions[y];
sharedpartitions[y]+=tmp;
}
}
__syncthreads();

Hist_dev_pre[2*thd]=sharedpartitions[2*thd];
Hist_dev_pre[2*thd+1]=sharedpartitions[2*thd+1];
}