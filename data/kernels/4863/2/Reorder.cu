#include "hip/hip_runtime.h"
#include "includes.h"

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
__device__ uint bfe(uint x, uint start, uint nbits)
{
uint bits;
asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
return bits;
}
__global__ void Reorder(long long* arrayofkeys, int* Hist_pre, int noofpartitions, long long size, long long* output)
{
register int thd = threadIdx.x;
register int bD = blockDim.x;
register int bI = blockIdx.x;
uint h,start,nbits;

start=0;
nbits=(uint)ceil(log2((float)noofpartitions));

long long thdindex= bD * bI + thd;

if(thdindex<size)
{
h=bfe(arrayofkeys[thdindex],start,nbits);
int offset=atomicAdd(&(Hist_pre[h]),1);
output[offset] = arrayofkeys[thdindex];

}

}