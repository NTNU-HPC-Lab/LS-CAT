#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
unsigned int i;
unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x*BLOCKSIZE);

if (ptr >= (unsigned long*) end_ptr) {
return;
}


for (i = 0;i < BLOCKSIZE/sizeof(unsigned long); i++){
ptr[i] =(unsigned long) & ptr[i];
}

return;
}