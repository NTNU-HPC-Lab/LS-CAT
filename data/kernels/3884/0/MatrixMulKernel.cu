#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Nathan Dunn
* CS-4370-90 Par. Prog. Many-Core GPUs
* Professor Liu
* 10-24-19
* Tiled Matrix Multiplication
*/

#define N 8 // size of the matrices to be multiplied
#define TILE_WIDTH 4 // size of the tiles

/**
* Computes the matrix multiplication on the CPU
* m - First matrix to be multiplied
* n - Second matrix to be multiplied
* p - Product of m and n
* width - Size of the matrices being operated upon
*/
__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width)
{
__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;  int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;

// Identify the row and column of the Pd element to work on
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;

double Pvalue = 0;
// Loop over the Md and Nd tiles required to compute the Pd element
for (int m = 0; m < Width/TILE_WIDTH; ++m){

// Collaborative loading of Md and Nd tiles into shared memory
ds_M[ty][tx] = d_M[Row*Width + m*TILE_WIDTH+tx];
ds_N[ty][tx] = d_N[Col+(m*TILE_WIDTH+ty)*Width];
__syncthreads();
for (int k = 0; k < TILE_WIDTH; ++k)
Pvalue += ds_M[ty][k] * ds_N[k][tx];
__syncthreads();
}
d_P[Row*Width+Col] = Pvalue;
}