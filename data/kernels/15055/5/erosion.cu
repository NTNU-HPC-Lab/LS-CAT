#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void erosion(uint8_t *inData, uint8_t *outData, int radiusX, int radiusY, int width, int height)
{
int gx = blockIdx.x * blockDim.x + threadIdx.x;
int gy = blockIdx.y * blockDim.y + threadIdx.y;

int x1 = gx - radiusX;
int x2 = gx + radiusX;
int y1 = gy - radiusY;
int y2 = gy + radiusY;

if (x1 < 0) {
x1 = 0;
} else if (x2 >= width) {
x2 = width - 1;
}

if (y1 < 0) {
y1 = 0;
} else if (y2 >= height) {
y2 = height - 1;
}

uint8_t minimum = 255;

for (int y = y1; y <= y2; ++y) {
for (int x = x1; x <= x2; ++x) {
minimum = min(minimum, inData[width * y + x]);
}
}

outData[width * gy + gx] = minimum;
}