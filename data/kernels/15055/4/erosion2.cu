#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void erosion2(uint8_t *inData, uint8_t *outData, int radiusX, int radiusY, int width, int height)
{
__shared__ uint8_t localData[TILE_SIZE * TILE_SIZE];

int tx = threadIdx.x;
int ty = threadIdx.y;
int gx = blockIdx.x * blockDim.x;
int gy = blockIdx.y * blockDim.y;

localData[TILE_SIZE * (radiusY + ty) + radiusX + tx] = inData[width * (gy + ty) + gx + tx];

int x1 = tx, y1 = ty, x2 = tx, y2 = ty;

if (tx == 0) {
x1 = max(0, gx - radiusX) - gx;
} else if (tx == blockDim.x - 1) {
x2 = min(width - 1 - gx, blockDim.x + radiusX - 1);
}

if (ty == 0) {
y1 = max(0, gy - radiusY) - gy;
} else if (ty == blockDim.y - 1) {
y2 = min(height - 1 - gy, blockDim.y + radiusY - 1);
}

__syncthreads();

for (int y = y1; y <= y2; ++y) {
for (int x = x1; x <= x2; ++x) {
localData[TILE_SIZE * (radiusY + y) + radiusX + x] = inData[width * (gy + y) + gx + x];
}
}

__syncthreads();

x1 = tx - radiusX;
x2 = tx + radiusX;
y1 = ty - radiusY;
y2 = ty + radiusY;

if (gx + x1 < 0) {
x1 = 0;
} else if (gx + x2 >= width) {
x2 = width - gx - 1;
}

if (gy + y1 < 0) {
y1 = 0;
} else if (gy + y2 >= height) {
y2 = height - gy - 1;
}

uint8_t minimum = 255;

for (int y = y1; y <= y2; ++y) {
for (int x = x1; x <= x2; ++x) {
minimum = min(minimum, localData[TILE_SIZE * (radiusY + y) + radiusX + x]);
}
}

outData[width * (gy + ty) + gx + tx] = minimum;
}