#include "hip/hip_runtime.h"
#include "includes.h"



#define PARTSIZE 4



__global__ void addSIMD(unsigned int *data1, unsigned int *data2)
{
int thread = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
unsigned int *ptr1 = data1 + thread;
unsigned int *ptr2 = data2 + thread;

*ptr1 = __vaddus4(*ptr1, *ptr2);
}