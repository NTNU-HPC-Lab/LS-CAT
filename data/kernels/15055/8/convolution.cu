#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution(uint8_t *inData, uint8_t *outData, int width, int height, float *kernel, int kwidth, int kheight, float ksum)
{
int gx = blockIdx.x * blockDim.x + threadIdx.x;
int gy = blockIdx.y * blockDim.y + threadIdx.y;

if (gx < width && gy < height) {
int rx = (kwidth - 1) / 2;
int ry = (kheight - 1) / 2;

float sum = 0.0;

for (int y = 0; y < kheight; ++y) {
int cy = max(0, min(height - 1, gy + y - ry));

for (int x = 0; x < kwidth; ++x) {
int cx = max(0, min(width - 1, gx + x - rx));
sum = fmaf((float) inData[cx + cy * width], kernel[x + y * kwidth], sum);
}
}

sum = max(0.0, min(255.0, sum));
outData[gx + gy * width] = (uint8_t) fdividef(sum, ksum);
}
}