#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void thresholdSIMD(unsigned int *data, unsigned int threshold)
{
int thread = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
unsigned int *ptr = data + thread;

*ptr = __vcmpgeu4(*ptr, threshold);
}