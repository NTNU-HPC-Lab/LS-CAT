#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply2(float* A, float* C, int size)
{
float sum = 0;
int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;

if(Col < size && Row < size) {
for (int k = 0; k < size; k++)
sum += A[k * size + Row] * A[k * size + Col];

C[Row * size + Col] = sum;
}
}