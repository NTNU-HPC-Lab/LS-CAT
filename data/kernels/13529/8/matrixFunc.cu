#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixFunc(float *F, int size)
{
#pragma unroll 16
for(int k = 0; k < 100; k++)
#pragma unroll 16
for(int i = 1; i < size; i++)
for(int j = 0; j < size - 1; j++)
F[i * size + j] = F[(i-1) * size + j + 1] + F[i * size + j + 1];
}