#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply1(float *A, float *C, int size) {
int Col = blockDim.y * blockIdx.y + threadIdx.y;
int Row = blockDim.x * blockIdx.x + threadIdx.x;


for(int k = 0; k < size; k++)
C[Row * size + Col] += A[k * size + Row] * A[k * size + Col];

}