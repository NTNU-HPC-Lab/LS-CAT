#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply3(float* A, float* C, int size) {

float CValue = 0;

int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

__shared__ float As[TILE_WIDTH][TILE_WIDTH];

for (int k = 0; k < (TILE_WIDTH + size - 1)/TILE_WIDTH; k++) {

if (k * TILE_WIDTH + threadIdx.x < size && Row < size)
As[threadIdx.y][threadIdx.x] = A[Row * size + k * TILE_WIDTH + threadIdx.x];
else
As[threadIdx.y][threadIdx.x] = 0.0;

if (k * TILE_WIDTH + threadIdx.y < size && Col < size)
As[threadIdx.y][threadIdx.x] = A[(k*TILE_WIDTH + threadIdx.y) * size + Col];
else
As[threadIdx.y][threadIdx.x] = 0.0;

__syncthreads();

for (int n = 0; n < TILE_WIDTH; ++n)
CValue += As[threadIdx.y][n] * As[n][threadIdx.x];

__syncthreads();
}

if (Row < size && Col < size)
C[((blockIdx.y * blockDim.y + threadIdx.y) * size) + (blockIdx.x*blockDim.x) + threadIdx.x] = CValue;
}