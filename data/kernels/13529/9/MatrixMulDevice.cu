#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulDevice( float *A, float *B, float *C, int *matrixSize)
{
int chunk = (*matrixSize) / gridDim.x;
int sum, i, k;

for(i = blockIdx.x * chunk; i < blockIdx.x * chunk + chunk - 1; i++) {
sum = 0;

for(k = 0; k < *matrixSize; k++) {
sum += A[i * *matrixSize + k] * B [k * *matrixSize + threadIdx.x];
}

C[i * *matrixSize + threadIdx.x] = sum;
}
}