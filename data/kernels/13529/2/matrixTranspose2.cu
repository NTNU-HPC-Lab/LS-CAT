#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define MAX_ARRAY_SIZE 1024
#define RANDOM_MAX  1000
#define TILE_DIM 16
#define BLOCK_ROWS 8
#define EPSILON 0.000001
#define NUM_BLOCKS (MAX_ARRAY_SIZE/TILE_DIM)

float A[MAX_ARRAY_SIZE][MAX_ARRAY_SIZE];
float C[MAX_ARRAY_SIZE][MAX_ARRAY_SIZE];

void serial();
void init_F();
int check();

__global__ void matrixTranspose2(const float *F, float *C)
{
__shared__ float tile[TILE_DIM][TILE_DIM];

int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
tile[threadIdx.y+j][threadIdx.x] = F[(y+j)*width + x];

__syncthreads();

x = blockIdx.y * TILE_DIM + threadIdx.x;
y = blockIdx.x * TILE_DIM + threadIdx.y;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
C[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}