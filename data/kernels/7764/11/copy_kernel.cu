#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(double *save, double *y) {
const int threadID = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
save[threadID] = y[threadID];
save[threadID + 1] = y[threadID + 1];
}