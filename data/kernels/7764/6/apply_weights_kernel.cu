#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_weights_kernel(double *g_out, int *g_in, double *g_ttmp) {
int val[2], test = 1;
double ttp_temp[2];
const int index = (blockIdx.x * blockDim.x + threadIdx.x) << 1;

val[0] = g_in[index];
val[1] = g_in[index + 1];
ttp_temp[0] = g_ttmp[index];
ttp_temp[1] = fabs(g_ttmp[index + 1]);

test = ttp_temp[0] < 0.0 ? 0 : 1;

g_out[index + 1] = (double) val[1] * ttp_temp[1];
ttp_temp[1] *= -g_ttp_inc[test];
g_out[index] = (double) val[0] * ttp_temp[1];
}