#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* count_d, float* randomnums)
{
int i;
double x,y,z;
int tid = blockDim.x * blockIdx.x + threadIdx.x;
i = tid;
int xidx = 0, yidx = 0;

xidx = (i+i);
yidx = (xidx+1);

x = randomnums[xidx];
y = randomnums[yidx];
z = ((x*x)+(y*y));

if (z<=1)
count_d[tid] = 1;
else
count_d[tid] = 0;
}