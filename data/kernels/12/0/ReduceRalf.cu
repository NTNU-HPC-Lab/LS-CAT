#include "hip/hip_runtime.h"
#include "includes.h"

//#define USE_DOUBLES

#ifdef USE_DOUBLES
typedef double my_float;
#else
typedef long long int my_float;
#endif

// convention   - any array without d_* is located on CPU
//              - any array with d_ is on device (GPU)

//const size_t NofS = 1<<20;
const size_t NofS = 1048576;
const size_t NofThreads = 1024;

//const size_t NofS=12;

__global__ void ReduceRalf(my_float *d_Array, my_float *d_ReducedArray, int N,int current)
{
int my_x = threadIdx.x+blockIdx.x*blockDim.x+current;
int tx=threadIdx.x;

__shared__ my_float sm[1024];
my_float cur=0.0;
if (my_x<N)
cur=d_Array[my_x];
if (my_x+blockDim.x*gridDim.x<N)
cur+=d_Array[my_x+blockDim.x*gridDim.x];

sm[tx]=cur;
for (int i=blockDim.x/2;i>0;i/=2)
{
__syncthreads();
if (tx<i)
sm[tx]=sm[tx]+sm[tx+i];
}
if (tx==0) d_ReducedArray[blockIdx.x]=sm[0];

}