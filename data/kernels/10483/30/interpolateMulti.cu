#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void interpolateMulti(int points, int noDims, int dimRes, float* surrogate, float* predictors, float* results) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < points) {
float *lower, *upper, *coeffs;
int *lowerInd;
lower = (float*)malloc((noDims)*sizeof(float));
upper = (float*)malloc((noDims)*sizeof(float));
coeffs = (float*)malloc(((int)pow(2,noDims-1))*sizeof(float));
lowerInd = (int*)malloc((noDims)*sizeof(float));

for (int jj = 0; jj < noDims; jj++) {
lower[jj] = surrogate[jj*dimRes];
upper[jj] = surrogate[(jj+1)*dimRes - 1];
lowerInd[jj] = (int)((dimRes-1)*(predictors[noDims*idx+jj] -
lower[jj])/(upper[jj] - lower[jj]));

if (lowerInd[jj] >= (dimRes-1)) {
lowerInd[jj] = dimRes-2;
} else if (lowerInd[jj] < 0){
lowerInd[jj] = 0;
}
}

// Let's interpolate
// Uppermost dimensions x value
float x0 = surrogate[lowerInd[0]];
float x1 = surrogate[lowerInd[0]+1];
float xd = (predictors[noDims*idx] - x0)/(x1-x0);

// First, assign the yvalues to the coefficients matrix
for (int jj = 0; jj < (int)pow(2,noDims-1); jj++) {
// Get the indices of the yvalues of the lower and upper bounding
// values on this dimension.
int idxL = dimRes*noDims;

for (int kk = 1; kk < noDims; kk++) {
int rem = ((int)(jj/((int)pow(2,noDims - kk - 1))) + 1) - 2*
(int)(((int)(jj/((int)pow(2,noDims - kk - 1))) + 1)/2);
if(rem > 0) {
idxL += lowerInd[kk]*(int)pow(dimRes,noDims - kk - 1);
} else {
idxL += (lowerInd[kk]+1)*(int)pow(dimRes,noDims - kk - 1);
}
}

int idxU = idxL + (lowerInd[0]+1)*(int)pow(dimRes,noDims-1);

idxL += lowerInd[0]*(int)pow(dimRes,noDims-1);

coeffs[jj] = surrogate[idxL]*(1 - xd) + surrogate[idxU]*xd;
}

// Now we work our way down the dimensions using our computed
// coefficients to get the interpolated value.
for (int jj = 1; jj < noDims; jj++) {
// Get the current dimension x value
x0 = surrogate[jj*dimRes + lowerInd[jj]];
x1 = surrogate[jj*dimRes + lowerInd[jj] + 1];
xd = (predictors[jj] - x0)/(x1-x0);

for (int kk = 0; kk < (int)pow(2,jj); kk++) {
int jump = (int)pow(2,noDims - jj - 2);
coeffs[kk] = coeffs[kk]*(1 - xd) + coeffs[kk + jump]*xd;
}
}

// Free variables
free(lowerInd);
free(coeffs);
free(upper);
free(lower);
// Output the result
results[idx] = coeffs[0];
}
}