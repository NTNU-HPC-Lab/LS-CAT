#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__device__ void solveLinearSystem(int dims, float *A, float *B, float *C) {
// First generate upper triangular matrix for the augmented matrix
float *swapRow;
swapRow = (float*)malloc((dims+1)*sizeof(float));

for (int ii = 0; ii < dims; ii++) {
C[ii] = B[ii];
}

for (int ii = 0; ii < dims; ii++) {
// Search for maximum in this column
float maxElem = fabsf(A[ii*dims+ii]);
int maxRow = ii;

for (int jj = (ii+1); jj < dims; jj++) {
if (fabsf(A[ii*dims+jj] > maxElem)) {
maxElem = fabsf(A[ii*dims+jj]);
maxRow = jj;
}
}

// Swap maximum row with current row if needed
if (maxRow != ii) {
for (int jj = ii; jj < dims; jj++) {
swapRow[jj] = A[jj*dims+ii];
A[jj*dims+ii] = A[jj*dims+maxRow];
A[jj*dims+maxRow] = swapRow[jj];
}

swapRow[dims] = C[ii];
C[ii] = C[maxRow];
C[maxRow] = swapRow[dims];
}

// Make all rows below this one 0 in current column
for (int jj = (ii+1); jj < dims; jj++) {
float factor = -A[ii*dims+jj]/A[ii*dims+ii];

// Work across columns
for (int kk = ii; kk < dims; kk++) {
if (kk == ii) {
A[kk*dims+jj] = 0.0;
} else {
A[kk*dims+jj] += factor*A[kk*dims+ii];
}
}

// Results vector
C[jj] += factor*C[ii];
}
}
free(swapRow);

// Solve equation for an upper triangular matrix
for (int ii = dims-1; ii >= 0; ii--) {
C[ii] = C[ii]/A[ii*dims+ii];

for (int jj = ii-1; jj >= 0; jj--) {
C[jj] -= C[ii]*A[ii*dims+jj];
}
}
}
__global__ void multiLocLinReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, int k, int* dataPoints, float *xvals, float *yvals, float *regression, float* xmins, float* xmaxes, float *dist, int *ind) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPoints) {
if (dataPoints[control] < 3) {
regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2)
+ control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*
noDims + idx] = 0.0;
} else {
// First, deconstruct the index into the index along each dimension
int *dimIdx;
dimIdx = (int*)malloc(noDims*sizeof(int));

int rem = idx;

for (int ii = 0; ii < noDims; ii++) {
int div = (int)(rem/pow(dimRes,noDims-ii-1));
dimIdx[ii] = div;
rem = rem - div*pow(dimRes,noDims-ii-1);
}

// Get the query point coordinates
float *xQ;
xQ = (float*)malloc(noDims*sizeof(float));

for (int ii = 0; ii < noDims; ii++) {
xQ[ii] = ((float)dimIdx[ii])*(xmaxes[ii] - xmins[ii])/(float)(
dimRes - 1) + xmins[ii];
}

// 1. First find the k nearest neighbours to the query point (already)
// computed prior).

// 2. Build the matrices used in the calculation
// A - Input design matrix
// B - Input known matrix
// C - Output matrix of coefficients
float *A, *B, *X;

A = (float*)malloc(pow(noDims+1,2)*sizeof(float));
B = (float*)malloc((noDims+1)*sizeof(float));
X = (float*)malloc((noDims+1)*sizeof(float));

// Bandwidth for kernel
float h = dist[noPoints*(k-1) + idx];

for (int ii = 0; ii <= noDims; ii++) {
// We will use a kernel and normalise by the distance of
// the furthest point of the nearest k neighbours.

// Initialise values to zero
B[ii] = 0.0;

for (int kk = 0; kk < k; kk++) {
float d = dist[noPoints*kk + idx];
// Gaussian kernel (Not used for now)
float z = exp(-(d/h)*(d/h)/2)/sqrt(2*M_PI);
// Epanechnikov kernel
//                    float z = 0.75*(1-pow(d/h,2));

if (ii == 0) {
B[ii] += yvals[ind[noPoints*kk + idx] - 1]*z;
} else {
B[ii] += yvals[ind[noPoints*kk + idx] - 1]*(xvals[(ind[noPoints
*kk + idx] - 1)*noDims + ii - 1] - xQ[ii-1])*z;
}
}

for (int jj = 0; jj <= noDims; jj++) {
A[jj*(noDims+1)+ii] = 0.0;

for (int kk = 0; kk < k; kk++) {
//                    float h = d_h[ind[kk]];
float d = dist[noPoints*kk + idx];
//                    For Gaussian kernel. Not used.
float z = exp(-(d/h)*(d/h)/2)/sqrt(2*M_PI);
//                        float z = 0.75*(1-pow(d/h,2));

if ((ii == 0) && (jj == 0)) {
A[jj*(noDims+1)+ii] += 1.0*z;
} else if (ii == 0) {
A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1
)*noDims + jj - 1] - xQ[jj - 1])*z;
} else if (jj == 0) {
A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1
)*noDims + ii - 1] - xQ[ii - 1])*z;
} else {
A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1
)*noDims + jj - 1] - xQ[jj-1])*(xvals[(ind[
noPoints*kk + idx] - 1)*noDims + ii - 1] - xQ[ii
- 1])*z;
}
}
}
}

// Solve the linear system using LU decomposition.
solveLinearSystem(noDims+1,A,B,X);

// 4. Compute the y value at the x point of interest using the just-
//    found regression coefficients. This is simply the y intercept we
//    just computed and save to the regression matrix.
regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2)
+ control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*
noDims + idx] = /*yvals[ind[idx] - 1]*/ X[0];

// Free memory
free(A);
free(B);
free(X);
free(xQ);
free(dimIdx);
}
}
}