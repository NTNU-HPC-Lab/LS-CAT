#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void buildGlobalQuadReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regCoeffs, float* xmins, float* xmaxes, float* regression) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPoints) {
// First deconstruct the index into the index along each dimension
int *dimIdx;
dimIdx = (int*)malloc(noDims*sizeof(int));

int rem = idx;

for (int ii = 0; ii < noDims; ii++) {
int div = (int)(rem/pow(dimRes,noDims-ii-1));
dimIdx[ii] = div;
rem = rem - div*pow(dimRes,noDims-ii-1);
}

// Get the query point coordinates
float *xQ;
xQ = (float*)malloc(noDims*sizeof(float));

for (int ii = 0; ii < noDims; ii++) {
xQ[ii] = ((float)dimIdx[ii])*(xmaxes[control*noDims + ii] -
xmins[control*noDims + ii])/(float)dimRes +
xmins[control*noDims + ii];
}

// Use the regression coefficients to compute the value at this query
// point
// Constant
float computed = regCoeffs[0];

// Linear Terms
for (int ii = 0; ii < noDims; ii++) {
computed += xQ[ii]*regCoeffs[ii+1];
}

// Quadratic and Interacting Terms
int counter = 0;
for (int ii = 0; ii < noDims; ii++) {
for (int jj = ii; jj < noDims; jj++) {
computed += xQ[ii]*xQ[jj]*regCoeffs[counter+1+noDims];
counter++;
}
}

// We know that a payoff cannot be greater than zero, so we adjust all
// conditional payoffs greater than zero to be zero.

if (computed >= 0.0) {
computed = 0.0;
}

regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2)
+ control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*
noDims + idx] = computed;

// Free memory
free(xQ);
free(dimIdx);
}
}