#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void computePathStates(int noPaths, int noDims, int nYears, int noControls, int year, float unitCost, float unitRevenue, int* controls, int noFuels, float *fuelCosts, float *uResults, float *uComposition, int noUncertainties, int *fuelIdx, int noCommodities, float* aars, float* totalPops, float* xin, int* currControls) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPaths) {

// 1. Adjusted population for each species
// We only take the highest flow's adjusted population as this is a
// measure of how damaging the road is. If we instead used the aar of
// the random control selected, we would get overlaps in the optimal
// control map.
for (int ii = 0; ii < noDims-1; ii++) {
//            xin[idx*noDims + ii] = totalPops[idx*(noDims-1)*(nYears+1) + year*
//                    (noDims-1) + ii]*aars[idx*(nYears+1)*noControls*(noDims-1)
//                    + year*noControls*(noDims-1) + ii*noControls + controls[
//                    idx*nYears + year]];
xin[idx*noDims + ii] = totalPops[idx*(noDims-1)*(nYears+1) + year*
(noDims-1) + ii]*aars[idx*(nYears+1)*noControls*(noDims-1)
+ year*noControls*(noDims-1) + ii*noControls + (noControls
- 1)];
}

// 2. Unit profit
float unitFuel = 0.0;
float orePrice = 0.0;

// Compute the unit fuel cost component
for (int ii = 0; ii < noFuels; ii++) {
unitFuel += fuelCosts[ii]*uResults[idx*(nYears+1)*noUncertainties +
(year)*noUncertainties + fuelIdx[ii]];
}
// Compute the unit revenue from ore
for (int ii = 0; ii < noCommodities; ii++) {
orePrice += uComposition[idx*nYears*noCommodities + (year)*
noCommodities + ii]*uResults[idx*(nYears+1)*noUncertainties +
(year)*noUncertainties + noFuels + ii];
}

xin[idx*noDims + noDims-1] = unitCost + unitFuel - unitRevenue*
orePrice;
currControls[idx] = controls[idx*nYears + year];

//        printf("%f %f\n",unitFuel,orePrice);
}
}