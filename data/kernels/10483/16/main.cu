#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "forwardPathKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int noPaths = 1;
int nYears = 1;
int noSpecies = 1;
int noPatches = 1;
int noControls = 1;
int noUncertainties = 1;
float timeStep = 1;
float *initPops = NULL;
hipMalloc(&initPops, XSIZE*YSIZE);
float *pops = NULL;
hipMalloc(&pops, XSIZE*YSIZE);
float *mmm = NULL;
hipMalloc(&mmm, XSIZE*YSIZE);
int *rowIdx = NULL;
hipMalloc(&rowIdx, XSIZE*YSIZE);
int *elemsPerCol = NULL;
hipMalloc(&elemsPerCol, XSIZE*YSIZE);
int maxElems = 1;
float *speciesParams = NULL;
hipMalloc(&speciesParams, XSIZE*YSIZE);
float *caps = NULL;
hipMalloc(&caps, XSIZE*YSIZE);
float *aars = NULL;
hipMalloc(&aars, XSIZE*YSIZE);
float *uncertParams = NULL;
hipMalloc(&uncertParams, XSIZE*YSIZE);
int *controls = NULL;
hipMalloc(&controls, XSIZE*YSIZE);
float *uJumps = NULL;
hipMalloc(&uJumps, XSIZE*YSIZE);
float *uBrownian = NULL;
hipMalloc(&uBrownian, XSIZE*YSIZE);
float *uJumpSizes = NULL;
hipMalloc(&uJumpSizes, XSIZE*YSIZE);
float *uJumpsSpecies = NULL;
hipMalloc(&uJumpsSpecies, XSIZE*YSIZE);
float *uBrownianSpecies = NULL;
hipMalloc(&uBrownianSpecies, XSIZE*YSIZE);
float *uJumpSizesSpecies = NULL;
hipMalloc(&uJumpSizesSpecies, XSIZE*YSIZE);
float *rgr = NULL;
hipMalloc(&rgr, XSIZE*YSIZE);
float *uResults = NULL;
hipMalloc(&uResults, XSIZE*YSIZE);
float *totalPops = NULL;
hipMalloc(&totalPops, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
forwardPathKernel<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noPatches,noControls,noUncertainties,timeStep,initPops,pops,mmm,rowIdx,elemsPerCol,maxElems,speciesParams,caps,aars,uncertParams,controls,uJumps,uBrownian,uJumpSizes,uJumpsSpecies,uBrownianSpecies,uJumpSizesSpecies,rgr,uResults,totalPops);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
forwardPathKernel<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noPatches,noControls,noUncertainties,timeStep,initPops,pops,mmm,rowIdx,elemsPerCol,maxElems,speciesParams,caps,aars,uncertParams,controls,uJumps,uBrownian,uJumpSizes,uJumpsSpecies,uBrownianSpecies,uJumpSizesSpecies,rgr,uResults,totalPops);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
forwardPathKernel<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noPatches,noControls,noUncertainties,timeStep,initPops,pops,mmm,rowIdx,elemsPerCol,maxElems,speciesParams,caps,aars,uncertParams,controls,uJumps,uBrownian,uJumpSizes,uJumpsSpecies,uBrownianSpecies,uJumpSizesSpecies,rgr,uResults,totalPops);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}