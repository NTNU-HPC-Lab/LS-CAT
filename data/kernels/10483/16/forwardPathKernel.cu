#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void forwardPathKernel(int noPaths, int nYears, int noSpecies, int noPatches, int noControls, int noUncertainties, float timeStep, float* initPops, float* pops, float*mmm, int* rowIdx, int* elemsPerCol, int maxElems, float* speciesParams, float* caps, float* aars, float* uncertParams, int* controls, float* uJumps, float* uBrownian, float* uJumpSizes, float* uJumpsSpecies, float* uBrownianSpecies, float* uJumpSizesSpecies, float* rgr, float* uResults, float* totalPops) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

// Only perform matrix multiplication sequentially for now. Later, if so
// desired, we can use dynamic parallelism because the card in the
// machine has CUDA compute compatability 3.5

if (idx < noPaths) {
// Initialise the population data at time t=0
for (int ii = 0; ii < noSpecies; ii++) {
float population = 0;
for (int jj = 0; jj < noPatches; jj++) {
pops[idx*(nYears+1)*noSpecies*noPatches + ii*noPatches + jj] =
initPops[jj];
population += pops[idx*(nYears+1)*noSpecies*noPatches + ii*
noPatches + jj];
}
totalPops[idx*(nYears+1)*noSpecies + ii] = population;

// The aars are computed in the next for loop.
}

// Carry over the initial value for all uncertainties
for (int ii = 0; ii < noUncertainties; ii++) {
uResults[idx*noUncertainties*(nYears+1) + ii] = uncertParams[ii*6];
}

float* grMean;
grMean = (float*)malloc(noSpecies*sizeof(float));

for (int ii = 0; ii < noSpecies; ii++) {
grMean[ii] = speciesParams[ii*8];
}

// All future time periods
for (int ii = 0; ii < nYears; ii++) {
// Control to pick
int control = controls[idx*nYears + ii];

for (int jj = 0; jj < noSpecies; jj++) {
totalPops[idx*(nYears+1)*noSpecies + (ii+1)*noSpecies + jj] =
0;

// Adjust the global growth rate mean for this species at this
// time step for this path.
float jump = (uJumpsSpecies[idx*noSpecies*nYears +
ii*noSpecies + jj] < speciesParams[jj*8 + 5]) ?
1.0f : 0.0f;
float meanP = speciesParams[jj*8 + 1];
float reversion = speciesParams[jj*8 + 4];

float brownian = uBrownianSpecies[idx*noSpecies*nYears +
ii*noSpecies + jj]*speciesParams[jj*8 + 2];
float jumpSize = uJumpSizesSpecies[idx*noSpecies*nYears
+ ii*noSpecies + jj]*pow(speciesParams[
jj*8 + 5],2) - pow(speciesParams[jj*8 + 5],2)/2;

grMean[jj] = grMean[jj] + reversion*(meanP - grMean[jj])*
timeStep + grMean[jj]*brownian + (exp(jumpSize) - 1)*
grMean[jj]*jump;

// Initialise temporary populations
float initialPopulation = 0.0f;

for (int kk = 0; kk < noPatches; kk++) {
initialPopulation += pops[idx*(nYears+1)*noSpecies*
noPatches + ii*noSpecies*noPatches + jj*noPatches
+ kk];
}

// For each patch, update the population for the next time
// period by using the movement and mortality matrix for the
// correct species/control combination. We use registers due
// to their considerably lower latency over global memory.
for (int kk = 0; kk < noControls; kk++) {
// Overall population at this time period
float totalPop = 0.0f;

int iterator = 0;
for (int ll = 0; ll < noPatches; ll++) {
// Population for this patch
float population = 0.0f;

// Transfer animals from each destination patch to
// this one for the next period.
for (int mm = 0; mm < elemsPerCol[(jj*noControls + kk)*
noPatches + ll]; mm++) {

float value = pops[idx*(nYears+1)*noSpecies*
noPatches + ii*noSpecies*noPatches + jj*
noPatches + rowIdx[iterator + (jj*
noControls + kk)*maxElems]]*mmm[iterator +
(jj*noControls + kk)*maxElems];

population += value;

iterator++;
}

totalPop += population;

// We only update the actual populations if we are in
// the control that was selected. Save the total
// population for the start of the next time period.
if (kk == control && ii < nYears) {
// Population growth based on a mean-reverting process
rgr[idx*noSpecies*noPatches*nYears + ii*noSpecies*
noPatches + jj*noPatches + ll] = grMean[jj]
+ rgr[idx*noSpecies*noPatches*nYears + ii*
noSpecies*noPatches + jj*noPatches + ll]*
speciesParams[jj*8 + 7];

float gr = rgr[idx*noSpecies*noPatches*nYears + ii*
noSpecies*noPatches + jj*noPatches + ll];

pops[idx*(nYears+1)*noSpecies*noPatches + (ii+1)*
noSpecies*noPatches + jj*noPatches + ll] =
population*(1.0f + gr*(caps[jj*noPatches +
ll] - population)/caps[jj*noPatches + ll]/
100.0);
totalPops[idx*noSpecies*(nYears+1) + (ii+1)*
noSpecies + jj] += pops[idx*(nYears+1)*
noSpecies*noPatches + (ii+1)*noSpecies*
noPatches + jj*noPatches + ll];
}
}
// Save AAR for this control at this time
aars[idx*(nYears+1)*noControls*noSpecies + ii*noControls*
noSpecies + jj*noControls + kk] = totalPop/
initialPopulation;
}
}

// Other uncertainties

for (int jj = 0; jj < noUncertainties; jj++) {
float jump = (uJumps[idx*noUncertainties*nYears +
ii*noUncertainties + jj] < uncertParams[jj*6 + 5]) ?
1.0f : 0.0f;

float curr = uResults[idx*noUncertainties*(nYears+1) +
ii*noUncertainties + jj];
float meanP = uncertParams[jj*6 + 1];
float reversion = uncertParams[jj*6 + 3];

float brownian = uBrownian[idx*noUncertainties*nYears +
ii*noUncertainties + jj]*uncertParams[jj*6 + 2];
float jumpSize = uJumpSizes[idx*noUncertainties*nYears +
ii*noUncertainties + jj]*pow(uncertParams[jj*6 + 4],2)
- pow(uncertParams[jj*6 + 4],2)/2;

// Save the value of the uncertainty for the next time period
uResults[idx*noUncertainties*(nYears+1)+(ii+1)*noUncertainties+jj]
= curr + reversion*(meanP - curr)*timeStep +
curr*brownian + (exp(jumpSize) - 1)*curr*jump;
}
}
free(grMean);
}
}