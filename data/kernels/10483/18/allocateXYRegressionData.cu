#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void allocateXYRegressionData(int noPaths, int noControls, int noDims, int nYears, float* speciesParams, int year, int* controls, float* xin, float *condExp, int *dataPoints, float *xvals, float *yvals) {

for (int ii = 0; ii < noControls; ii++) {
dataPoints[ii] = 0;
}

//    // For each path
for (int ii = 0; ii < noPaths; ii++) {
if (controls[ii] >= noControls) {
printf("Invalid control %d\n",controls[ii]);
}

// NOT CHECKING VALIDITY
//        yvals[noPaths*controls[ii] + dataPoints[controls[ii]]] = condExp[(
//                year + 1)*noPaths + ii];

//        // Save the input dimension values to the corresponding data group
//        for (int jj = 0; jj < noDims; jj++) {
//            xvals[controls[ii]*noPaths*noDims + jj*noPaths + dataPoints[
//                    controls[ii]]] = xin[ii*noDims + jj];
//        }

////        printf("%6d | %3d: %6.0f %15.0f %15.0f\n",ii,controls[ii],xin[ii*noDims],
////                xin[ii*noDims + 1],yvals[noPaths*controls[ii] + dataPoints[controls[ii]]]);

//        // Increment the number of data points for this control
//        dataPoints[controls[ii]] += 1;

// CHECKING
// First check that the path is in-the-money. If it isn't we do not use
// it
bool valid = true;
for (int jj = 0; jj < (noDims-1); jj++) {
if (xin[ii*noDims + jj] < speciesParams[8*jj + 3]) {
valid = false;
break;
}
}

if (valid || controls[ii] == 0) {
// Save the conditional expectation
yvals[noPaths*controls[ii] + dataPoints[controls[ii]]] = condExp[(
year + 1)*noPaths + ii];

// Save the input dimension values to the corresponding data group
for (int jj = 0; jj < noDims; jj++) {
xvals[controls[ii]*noPaths*noDims + jj*noPaths + dataPoints[
controls[ii]]] = xin[ii*noDims + jj];
}

// Increment the number of data points for this control
dataPoints[controls[ii]] += 1;
}
}
}