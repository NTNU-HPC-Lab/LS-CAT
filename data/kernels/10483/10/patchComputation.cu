#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void patchComputation(int noCandidates, int W, int H, int skpx, int skpy, int xres, int yres, float subPatchArea, float xspacing, float yspacing, float capacity, int uniqueRegions, const int* labelledImage, const float* pops, float* results) {

// Get global index of thread
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noCandidates) {
// Dimensions arranged as X->Y->R
int rem = idx;
int blockIdxY = (int)(idx/(xres*uniqueRegions));
rem = rem - blockIdxY*(xres*uniqueRegions);
int blockIdxX = (int)(rem/uniqueRegions);
rem = rem - blockIdxX*(uniqueRegions);
// Valid region numbering starts at 1, not 0
int regionNo = rem + 1;

int blockSizeX;
int blockSizeY;

if ((blockIdxX+1)*skpx <= H) {
blockSizeX = skpx;
} else {
blockSizeX = H-blockIdxX*skpx;
}

if ((blockIdxY+1)*skpy <= W) {
blockSizeY = skpy;
} else {
blockSizeY = W-blockIdxY*skpy;
}

// Iterate through each sub patch for this large grid cell
float area = 0.0f;
float cap = 0.0f;
float pop = 0.0f;
float cx = 0.0f;
float cy = 0.0f;

for (int ii = 0; ii < blockSizeX; ii++) {
for (int jj = 0; jj < blockSizeY; jj++) {
int xCoord = blockIdxX*skpx+ii;
int yCoord = blockIdxY*skpy+jj;

area += (float)(labelledImage[xCoord + yCoord*W] == regionNo);
}
}

if (area > 0) {
for (int ii = 0; ii < blockSizeX; ii++) {
for (int jj = 0; jj < blockSizeY; jj++) {
int xCoord = blockIdxX*skpx+ii;
int yCoord = blockIdxY*skpy+jj;

if (labelledImage[xCoord + yCoord*W] == regionNo) {
pop += (float)pops[xCoord + yCoord*W];
cx += ii;
cy += jj;
}
}
}
cx = xspacing*(cx/area + blockIdxX*skpx);
cy = yspacing*(cy/area + blockIdxY*skpy);
area = area*subPatchArea;
cap = area*capacity;
}

// Store results to output matrix
results[5*idx] = area;
results[5*idx+1] = cap;
results[5*idx+2] = pop;
results[5*idx+3] = cx;
results[5*idx+4] = cy;
}
}