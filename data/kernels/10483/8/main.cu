#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "pathAdjacencyKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int noTransitions = 1;
int noSegments = 1;
float *XY1 = NULL;
hipMalloc(&XY1, XSIZE*YSIZE);
float *XY2 = NULL;
hipMalloc(&XY2, XSIZE*YSIZE);
float *X4_X3 = NULL;
hipMalloc(&X4_X3, XSIZE*YSIZE);
float *Y4_Y3 = NULL;
hipMalloc(&Y4_Y3, XSIZE*YSIZE);
float *X2_X1 = NULL;
hipMalloc(&X2_X1, XSIZE*YSIZE);
float *Y2_Y1 = NULL;
hipMalloc(&Y2_Y1, XSIZE*YSIZE);
int *adjacency = NULL;
hipMalloc(&adjacency, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
pathAdjacencyKernel<<<gridBlock,threadBlock>>>(noTransitions,noSegments,XY1,XY2,X4_X3,Y4_Y3,X2_X1,Y2_Y1,adjacency);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
pathAdjacencyKernel<<<gridBlock,threadBlock>>>(noTransitions,noSegments,XY1,XY2,X4_X3,Y4_Y3,X2_X1,Y2_Y1,adjacency);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
pathAdjacencyKernel<<<gridBlock,threadBlock>>>(noTransitions,noSegments,XY1,XY2,X4_X3,Y4_Y3,X2_X1,Y2_Y1,adjacency);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}