#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void pathAdjacencyKernel(int noTransitions, int noSegments, float* XY1, float* XY2, float* X4_X3, float* Y4_Y3, float* X2_X1, float* Y2_Y1, int* adjacency) {

int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int idx = blockId * blockDim.x + threadIdx.x;

if (idx < noTransitions*noSegments) {
int seg1 = idx/noSegments;
int seg2 = idx - seg1*noSegments;

float Y1_Y3 = XY1[seg1 + noTransitions] - XY2[seg2 + noSegments];
float X1_X3 = XY1[seg1] - XY2[seg2];

float numa = X4_X3[seg2]*Y1_Y3 - Y4_Y3[seg2]*X1_X3;
float numb = X2_X1[seg1]*Y1_Y3 - Y2_Y1[seg1]*X1_X3;
float deno = Y4_Y3[seg2]*X2_X1[seg1] - X4_X3[seg2]*Y2_Y1[seg1];

float u_a = numa/deno;
float u_b = numb/deno;

adjacency[idx] = (int)((u_a >= 0.0) && (u_a <= 1.0) && (u_b >= 0.0)
&& (u_b <= 1.0));
}
}