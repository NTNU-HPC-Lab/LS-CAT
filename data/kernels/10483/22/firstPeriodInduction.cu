#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void firstPeriodInduction(int noPaths, int nYears, int noSpecies, int noControls, float timeStep, float unitCost, float unitRevenue, float rrr, int noFuels, int noCommodities, float* Q, float* fuelCosts, float* totalPops, float* speciesParams, int* controls, float* aars, float* uComposition, float* uResults, int* fuelIdx, float* condExp, int* optCont, float* stats) {

float *payoffs, *dataPoints;
payoffs = (float*)malloc(noControls*sizeof(float));
dataPoints = (float*)malloc(noControls*sizeof(float));
bool* valid;
valid = (bool*)malloc(noControls*sizeof(bool));

float unitFuel = 0.0;
float orePrice = 0.0;

// Compute the unit fuel cost component
for (int ii = 0; ii < noFuels; ii++) {
unitFuel += fuelCosts[ii]*uResults[fuelIdx[ii]];
}
// Compute the unit revenue from ore
for (int ii = 0; ii < noCommodities; ii++) {
orePrice += uComposition[ii]*uResults[noFuels + ii];
}

for (int ii = 0; ii < noControls; ii++) {
dataPoints[ii] = 0.0;
payoffs[ii] = 0.0;
}

// Now get the average payoff across all paths of the same control for
// each control

for (int ii = 0; ii < noPaths; ii++) {
int control = controls[ii*nYears];

payoffs[control] += condExp[ii+noPaths];
dataPoints[control]++;
}

for (int ii = 0; ii < noControls; ii++) {
// Compute the single period financial payoff for each control
// for this period and the adjusted profit. If any adjusted
// population is below the threshold, then the payoff is
// invalid.
if (dataPoints[ii] > 0) {
payoffs[ii] = payoffs[ii]/(dataPoints[ii]*(1+rrr*timeStep/
100));
} else {
break;
}

valid[ii] = true;
for (int jj = 0; jj < noSpecies; jj++) {
float adjPop = totalPops[jj]*aars[jj*noControls + ii];

// Zero flow control is always valid
if (adjPop < speciesParams[noSpecies*jj + 3] && ii > 0) {
valid[ii] = false;
break;
}
}

// Compute the payoff for the control if valid.
if (valid[ii]) {
// Now compute the overall period profit for this control
// given the prevailing stochastic factors (undiscounted).
payoffs[ii] += Q[ii]*(unitCost + unitFuel - unitRevenue*
orePrice);

// Take care of regression anomalies
if (payoffs[ii] > 0) {
payoffs[ii] = 0.0;
}
} else {
payoffs[ii] = NAN;
}
}

//    printf("Pop: %6.2f %6.2f %6.2f\n", totalPops[0]*aars[0], totalPops[0]*aars[1],totalPops[0]*aars[2]);

// The optimal value is the one with the lowest net present cost.
// As the zero flow rate option is always available, we can
// initially set the optimal control to this before checking the
// other controls.
float bestExp = payoffs[0];
int bestCont = 0;

for (int ii = 1; ii < noControls; ii++) {
if (isfinite(payoffs[ii])) {
if (payoffs[ii] < bestExp) {
bestExp = payoffs[ii];
bestCont = ii;
}
}
}

// Assign the optimal control and payoff to all paths at time period 0

// Standard deviation
stats[2] = 0;

// Assign values and prepare standard deviation
for (int ii = 0; ii < noPaths; ii++) {
condExp[ii] = bestExp;
optCont[ii] = bestCont;

if (controls[ii*nYears] == bestCont) {
stats[2] += (condExp[ii+noPaths] - payoffs[bestCont])*(condExp[ii
+noPaths] - payoffs[bestCont]);
}
}

stats[0] = condExp[0];
stats[1] = (float)optCont[0];
stats[2] = sqrt(stats[2]/(dataPoints[bestCont]*(1+rrr/(100*timeStep))));

free(valid);
free(payoffs);
free(dataPoints);
}