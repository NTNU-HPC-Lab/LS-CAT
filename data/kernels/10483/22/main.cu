#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "firstPeriodInduction.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int noPaths = 1;
int nYears = 1;
int noSpecies = 1;
int noControls = 1;
float timeStep = 1;
float unitCost = 1;
float unitRevenue = 1;
float rrr = 1;
int noFuels = 1;
int noCommodities = 1;
float *Q = NULL;
hipMalloc(&Q, XSIZE*YSIZE);
float *fuelCosts = NULL;
hipMalloc(&fuelCosts, XSIZE*YSIZE);
float *totalPops = NULL;
hipMalloc(&totalPops, XSIZE*YSIZE);
float *speciesParams = NULL;
hipMalloc(&speciesParams, XSIZE*YSIZE);
int *controls = NULL;
hipMalloc(&controls, XSIZE*YSIZE);
float *aars = NULL;
hipMalloc(&aars, XSIZE*YSIZE);
float *uComposition = NULL;
hipMalloc(&uComposition, XSIZE*YSIZE);
float *uResults = NULL;
hipMalloc(&uResults, XSIZE*YSIZE);
int *fuelIdx = NULL;
hipMalloc(&fuelIdx, XSIZE*YSIZE);
float *condExp = NULL;
hipMalloc(&condExp, XSIZE*YSIZE);
int *optCont = NULL;
hipMalloc(&optCont, XSIZE*YSIZE);
float *stats = NULL;
hipMalloc(&stats, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
firstPeriodInduction<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noControls,timeStep,unitCost,unitRevenue,rrr,noFuels,noCommodities,Q,fuelCosts,totalPops,speciesParams,controls,aars,uComposition,uResults,fuelIdx,condExp,optCont,stats);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
firstPeriodInduction<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noControls,timeStep,unitCost,unitRevenue,rrr,noFuels,noCommodities,Q,fuelCosts,totalPops,speciesParams,controls,aars,uComposition,uResults,fuelIdx,condExp,optCont,stats);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
firstPeriodInduction<<<gridBlock,threadBlock>>>(noPaths,nYears,noSpecies,noControls,timeStep,unitCost,unitRevenue,rrr,noFuels,noCommodities,Q,fuelCosts,totalPops,speciesParams,controls,aars,uComposition,uResults,fuelIdx,condExp,optCont,stats);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}