#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void computeStateMinMax(int noControls, int noDims, int noPaths, int* dataPoints, float* xvals, float* xmins, float* xmaxes) {

for (int ii = 0; ii < noControls; ii++) {
float *xmin, *xmax;
xmin = (float*)malloc(noDims*sizeof(float));
xmax = (float*)malloc(noDims*sizeof(float));

if (ii == 0 || dataPoints[ii] > (noDims+1)) {
for (int jj = 0; jj < noDims; jj++) {
xmin[jj] = xvals[ii*noDims*noPaths + jj*noPaths];
xmax[jj] = xmin[jj];
}

for (int jj = 0; jj < noDims; jj++) {
for (int kk = 0; kk < dataPoints[ii]; kk++) {
float xtemp = xvals[ii*noDims*noPaths + jj*noPaths + kk];
if (xmin[jj] > xtemp) {
xmin[jj] = xtemp;
} else if (xmax[jj] < xtemp) {
xmax[jj] = xtemp;
}
}
}

//        for (int jj = 0; jj < noDims; jj++) {
//            xmin[jj] = xvals[ii*noDims*noPaths + jj];
//            xmax[jj] = xmin[jj];
//        }

//        for (int jj = 0; jj < dataPoints[ii]; jj++) {
//            for (int kk = 0; kk < noDims; kk ++) {
//                float xtemp = xvals[ii*noDims*noPaths + jj*noDims + kk];
//                if (xmin[kk] > xtemp) {
//                    xmin[kk] = xtemp;
//                } else if (xmax[kk] < xtemp) {
//                    xmax[kk] = xtemp;
//                }
//            }
//        }

for (int jj = 0; jj < noDims; jj++) {
xmins[ii*noDims + jj] = xmin[jj];
xmaxes[ii*noDims + jj] = xmax[jj];
//            printf("Control %d: Xmin = %f Xmax = %f\n",ii,xmin[jj],xmax[jj]);
}
} else {
for (int jj = 0; jj < noDims; jj++) {
xmins[ii*noDims + jj] = xmins[jj];
xmaxes[ii*noDims + jj] = xmaxes[jj];
}
}

free(xmin);
free(xmax);
}

for (int ii = 0; ii < noDims; ii++) {
xmins[noControls*noDims + ii] = xmins[ii];
xmaxes[noControls*noDims + ii] = xmaxes[ii];
}

for (int ii = 1; ii < noControls; ii++) {
for (int jj = 0; jj < noDims; jj++) {
float xtemp = xmins[ii*noDims + jj];
if (xmins[noControls*noDims + jj] > xtemp) {
xmins[noControls*noDims + jj] = xtemp;
}

xtemp = xmaxes[ii*noDims + jj];
if (xmaxes[noControls*noDims + jj] < xtemp) {
xmaxes[noControls*noDims + jj] = xtemp;
}
}
}
}