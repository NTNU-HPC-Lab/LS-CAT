#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void buildGlobalLinReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regCoeffs, float* xmins, float* xmaxes, float* regression) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPoints) {
// First, deconstruct the index into the index along each dimension
int *dimIdx;
dimIdx = (int*)malloc(noDims*sizeof(int));

int rem = idx;

for (int ii = 0; ii < noDims; ii++) {
int div = (int)(rem/pow(dimRes,noDims-ii-1));
dimIdx[ii] = div;
rem = rem - div*pow(dimRes,noDims-ii-1);
}

// Get the query point coordinates
float *xQ;
xQ = (float*)malloc(noDims*sizeof(float));

for (int ii = 0; ii < noDims; ii++) {
xQ[ii] = ((float)dimIdx[ii])*(xmaxes[control*noDims + ii] -
xmins[control*noDims + ii])/(float)dimRes +
xmins[control*noDims + ii];
}

// Use the regression coefficients to compute the value at this query
// point
float computed = regCoeffs[0];

for (int ii = 0; ii < noDims; ii++) {
computed += xQ[ii]*regCoeffs[ii+1];
}

if (computed >= 0) {
computed = 0;
}

regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2)
+ control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*
noDims + idx] = computed;

// Free memory
free(xQ);
free(dimIdx);
}
}