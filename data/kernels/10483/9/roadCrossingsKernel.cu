#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void roadCrossingsKernel(int rows, int segs, int* adjacency, int* cross) {

int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < rows) {
cross[idx] = 0;

for (int ii = 0; ii < segs; ii++) {
cross[idx] += adjacency[idx*segs + ii];
}
}
}