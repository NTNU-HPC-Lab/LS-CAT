#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void printAverages(int nYears, int noSpecies, int noControls, int noPaths, float* totalPops, float* aars) {
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < nYears) {
float* totals, *aar;
totals = (float*)malloc(noSpecies*sizeof(float));

aar = (float*)malloc(noSpecies*noControls*sizeof(float));

for (int ii = 0; ii < noSpecies; ii++) {
totals[ii] = 0.0f;
for (int kk = 0; kk < noControls; kk++) {
aar[ii*noControls + kk] = 0;
}
}

for (int ii = 0; ii < noPaths; ii++) {
for (int jj = 0; jj < noSpecies; jj++) {
totals[jj] += totalPops[ii*noSpecies*(nYears+1) + (idx+1)*
noSpecies + jj];
for (int kk = 0; kk < noControls; kk++) {
aar[jj*noControls + kk] += aars[ii*(nYears+1)*noControls*
noSpecies + idx*noControls*noSpecies + jj*
noControls + kk];
}
}
}

for (int ii = 0; ii < noSpecies; ii++) {
totals[ii] = totals[ii]/(float)noPaths;
for (int jj = 0; jj < noControls; jj++) {
aar[ii*noControls + jj] = aar[ii*noControls + jj]/(float)
noPaths;
}
}

printf("Year: %d Total: %f C1: %f C2: %f C3: %f\n", idx,totals[0],aar[0],aar[1],aar[2]);

free(totals);
free(aar);
}
}