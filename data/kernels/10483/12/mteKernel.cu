#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void mteKernel(int noPaths, int nYears, int noPatches, float timeStep, float* rgr, float* brownians, float* jumpSizes, float* jumps, float* speciesParams, float *initPops, float* caps, float*mmm, int* rowIdx, int* elemsPerCol, float* pathPops, float* eps) {
// Global index for finding the thread number
int ii = blockIdx.x*blockDim.x + threadIdx.x;

// Only perform matrix multiplication sequentially for now. Later, if
// so desired, we can use dynamic parallelism because the card in the
// machine has CUDA compute capability 3.5
if (ii < noPaths) {
//extern __shared__ float s[];

// Initialise the prevailing population vector
for (int jj = 0; jj < noPatches; jj++) {
pathPops[(ii*2)*noPatches+jj] = initPops[jj];
}

float grMean = speciesParams[0];

for (int jj = 0; jj < nYears; jj++) {
// Movement and mortality. This component is very slow without
// using shared memory. As we do not know the size of the patches
// at compile time, we need to be careful how much shared memory we
// allocate. For safety, we assume that we will have less than
// 64KB worth of patch data in the mmm matrix. Using single
// precision floating point numbers, this means that we can only
// have up to 8,000 patches. As this number is extremely large, we
// set a limit outside this routine to have at most 300 patches.
for (int kk = 0; kk < noPatches; kk++) {
pathPops[(ii*2+1)*noPatches+kk] = 0.0;
}

int iterator = 0;
for (int kk = 0; kk < noPatches; kk++) {
for (int ll = 0; ll < elemsPerCol[kk]; ll++) {
pathPops[(ii*2+1)*noPatches+kk] += pathPops[(ii*2)*
noPatches+rowIdx[iterator]]*mmm[iterator];
iterator++;
}
}

// UPDATE: NEED TO IMPLEMENT SHARED MEMORY AS WELL

// DEPRECATED - TO BE DELETED AT LATER STAGE
// Load the correct slice of the mmm matrix for each
// destination patch. Use the thread index as a helper to do
// this. Wait for all information to be loaded in before
// proceeding. We need to tile the mmm matrix here to obtain
// a sufficient speed up.

//            for (int kk = 0; kk < noTiles; kk++) {
//                int currDim = tileDim;

//                if (threadIdx.x < noPatches) {
//                    // First, allocate the memory for this tile
//                    if (kk == noTiles-1) {
//                        currDim = (int)(noTiles*tileDim == noPatches) ?
//                                (int)tileDim : (int)(noPatches - kk*tileDim);
//                    }

//                    for (int ll = 0; ll < currDim; ll++) {
//                        s[ll*noPatches + threadIdx.x] = mmm[kk*noPatches*
//                                tileDim + ll*noPatches + threadIdx.x];
//                    }
//                }
//                __syncthreads();

//                // Now increment the populations for this path
//                for (int kk = 0; kk < currDim; kk++) {
//                    for (int ll = 0; ll < noPatches; ll++) {
//                        pathPops[(ii*2+1)*noPatches+kk] += pathPops[(ii*2)*
//                                noPatches+ll]*s[kk*noPatches + ll];
//                    }
//                }
//            }

//            for (int kk = 0; kk < noPatches; kk++) {
//                for (int ll = 0; ll < noPatches; ll++) {
////                    pathPops[(ii*2+1)*noPatches+kk] += pathPops[(ii*2)*
////                            noPatches+ll]*s[ll];
//                    pathPops[(ii*2+1)*noPatches+kk] += pathPops[(ii*2)*
//                            noPatches+ll]*mmm[kk*noPatches+ll];
//                }
//            }

//            matrixMultiplicationKernel<<<noBlocks,noThreadsPerBlock>>>(pathPops
//                    + (ii*2)*noPatches, mmm, pathPops + (ii*2+1)*noPatches, 1,
//                    noPatches, noPatches);
//            hipDeviceSynchronize();
//            __syncthreads();

// Natural birth and death

// Adjust the global growth rate mean for this species at this
// time step for this path.
float jump = (jumps[ii*nYears + jj] < speciesParams[6]) ? 1.0f :
0.0f;
float meanP = speciesParams[1];
float reversion = speciesParams[4];

float brownian = brownians[ii*nYears + jj]*speciesParams[2];
float jumpSize = jumpSizes[ii*nYears + jj]*pow(speciesParams[5],2)
- pow(speciesParams[5],2)/2;

grMean = grMean + reversion*(meanP - grMean)*timeStep + grMean
*brownian + (exp(jumpSize) - 1)*grMean*jump;

for (int kk = 0; kk < noPatches; kk++) {
float gr = speciesParams[7]*rgr[ii*(nYears*noPatches) + jj*
noPatches + kk]*grMean + grMean;
pathPops[(ii*2)*noPatches+kk] = pathPops[(ii*2+1)*noPatches+kk]
*(1.0f + gr*(caps[kk]-pathPops[(ii*2+1)*noPatches+kk])/
caps[kk]);
}
}

eps[ii] = 0.0f;
for (int jj = 0; jj < noPatches; jj++) {
eps[ii] += pathPops[(ii*2+1)*noPatches+jj];
}
}
}