#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void rovCorrection(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regression) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPoints) {
float currVal = regression[year*noControls*(dimRes*noDims +
(int)pow(dimRes,noDims)*2) + control*(dimRes*noDims +
(int)pow(dimRes,noDims)*2) + dimRes*noDims + idx];

// The surrogate value cannot be greater than zero by definition
if (currVal > 0) {
regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,
noDims)*2) + control*(dimRes*noDims + (int)pow(dimRes,
noDims)*2) + dimRes*noDims + idx] = 0.0;
}
}
}