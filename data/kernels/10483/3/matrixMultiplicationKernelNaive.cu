#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernelNaive(const float* A, const float* B, float* C, int a, int b, int c, int d) {

int ROW = blockIdx.y*blockDim.y+threadIdx.y;
int COL = blockIdx.x*blockDim.x+threadIdx.x;

float tmpSum = 0.0f;

if (ROW < a && COL < d) {
// each thread computes one element of the block sub-matrix
for (int ii = 0; ii < b; ii++) {
tmpSum += A[ROW * b + ii] * B[ii * b + COL];
}
}
C[ROW * a + COL] = tmpSum;
}