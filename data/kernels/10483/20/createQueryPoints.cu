#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void createQueryPoints(int noPoints, int noDims, int dimRes, int control, int noControls, int year, float* xmins, float* xmaxes, float* regression, float* queryPts) {

// Global thread index
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPoints) {

// First, deconstruct the index into the index along each dimension
int *dimIdx;
dimIdx = (int*)malloc(noDims*sizeof(int));

int rem = idx;

for (int ii = 0; ii < noDims; ii++) {
int div = (int)(rem/pow(dimRes,noDims-ii-1));
dimIdx[ii] = div;
rem = rem - div*pow(dimRes,noDims-ii-1);
}

// We use the highest and lowest x values for each dimension
// among ALL the controls, not just for this control

// Get the query point coordinates
for (int ii = 0; ii < noDims; ii++) {
//            queryPts[idx + ii*noPoints] = ((float)dimIdx[ii])*(xmaxes[
//                    control*noDims + ii] - xmins[control*noDims + ii])/(
//                    float)(dimRes-1) + xmins[control*noDims + ii];
queryPts[idx + ii*noPoints] = ((float)dimIdx[ii])*(xmaxes[
noControls*noDims + ii] - xmins[noControls*noDims +
ii])/(float)(dimRes-1) + xmins[noControls*noDims +
ii];

// Save the X value for the query point
regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,
noDims)*2) + control*(dimRes*noDims + (int)pow(dimRes,
noDims)*2) + ii*dimRes + dimIdx[ii]] = queryPts[idx + ii*
noPoints];
}

free(dimIdx);
}
}