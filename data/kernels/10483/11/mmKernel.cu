#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void mmKernel(float* popsIn, float* popsOut, float* mmm, int patches) {
int ii = threadIdx.x;

if (ii < patches) {
extern __shared__ float s[];

s[ii] = 0.0;

for (int jj = 0; jj < patches; jj++) {
s[ii] += popsIn[ii]*mmm[ii*patches + jj];
}
__syncthreads();

popsOut[ii] = s[ii];
}
}