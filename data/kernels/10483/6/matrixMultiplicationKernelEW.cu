#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernelEW(const float* A, const float* B, float* C, int a, int b) {

int ROW = blockIdx.y*blockDim.y+threadIdx.y;
int COL = blockIdx.x*blockDim.x+threadIdx.x;

if (ROW < a && COL < b) {
C[ROW * a + COL] = A[ROW * b + COL]*B[ROW * b + COL];
}
}