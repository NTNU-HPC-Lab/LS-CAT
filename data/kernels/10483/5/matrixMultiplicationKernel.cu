#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernel(float *A, float* B, float* C, int a, int b, int d) {

// Block index
int bx = blockIdx.x;
int by = blockIdx.y;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

int ROW = by*blockDim.y+ty;
int COL = bx*blockDim.x+tx;

// First check if the thread exceeds the matrix dimensions
if (ROW < a && COL < d) {

// Declaration of the shared memory array As used to store the sub-
// matrix of A
__shared__ float As[BLOCK_SIZE * BLOCK_SIZE];
__shared__ float As2[BLOCK_SIZE * BLOCK_SIZE];

float *prefetch = As;
float *prefetch2 = As2;

// Declaration of the shared memory array Bs used to
// store the sub-matrix of B
// __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

float cv[BLOCK_SIZE];

for (int ii = 0; ii < BLOCK_SIZE; ii++) {
cv[ii] = 0;
}

// Index of the first sub-matrix of A processed by the block
int aBegin = a * BLOCK_SIZE * by;

// Index of the last sub-matrix of A processed by the block
int aEnd   = aBegin + a - 1;

// Step size used to iterate through the sub-matrices of A
int aStep  = BLOCK_SIZE;

// Index of the first sub-matrix of B processed by the block
int bBegin = BLOCK_SIZE * VECTOR_SIZE * bx;

// Step size used to iterate through the sub-matrices of B
int bStep  = BLOCK_SIZE * d;

int cBegin = d * BLOCK_SIZE * by + VECTOR_SIZE * BLOCK_SIZE * bx;

// Csub is used to store the element of the block sub-matrix
// that is computed by the thread
// float Csub = 0;
float *Ap = &A[aBegin + a * ty +tx];
float *ap = &prefetch[ty + BLOCK_SIZE * tx];
#pragma unroll
for(int ii = 0; ii < BLOCK_SIZE; ii+=4){
ap[ii] = Ap[a * ii];
}
__syncthreads();

// Loop over all the sub-matrices of A and B
// required to compute the block sub-matrix
for (int a = aBegin, b = bBegin;
a <= aEnd;
a += aStep, b += bStep) {

// Load the matrices from device memory
// to shared memory; each thread loads
// one element of each matrix
Ap = &A[a + aStep + a * ty +tx];
float *ap2 = &prefetch2[ty + BLOCK_SIZE * tx];
#pragma unroll
for(int ii = 0; ii < BLOCK_SIZE; ii+=4){
ap2[ii] = Ap[b * ii];
}

ap = &prefetch[0];
float *bp = &B[b + BLOCK_SIZE * ty + tx];

#pragma unroll
for (int ii = 0; ii < BLOCK_SIZE; ii++) {
float bv = bp[0];
for (int jj = 0; jj < BLOCK_SIZE; jj++) {
cv[jj] += ap[jj]*bv;
ap += BLOCK_SIZE;
bp += d;
}
}

// Synchronize to make sure the matrices are loaded
__syncthreads();

// swap As and As2
float *prefetch_temp = prefetch;
prefetch = prefetch2;
prefetch2 = prefetch_temp;
}

// Write the block sub-matrix to device memory;
// each thread writes one element
float *Cp = &C[cBegin];
Cp += BLOCK_SIZE * ty + tx;
int cStep = d;
#pragma unroll
for(int ii=0; ii<BLOCK_SIZE; ii++){
Cp[0] = cv[ii]; Cp += cStep;
}
}
}