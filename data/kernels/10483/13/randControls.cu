#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void randControls(int noPaths, int nYears, int noControls, float* randCont, int* control) {
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < noPaths*nYears) {
control[idx] = (int)(randCont[idx]*noControls);
if (control[idx] == noControls) {
control[idx]--;
}
}
}