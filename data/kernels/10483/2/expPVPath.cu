#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void expPVPath(const int noPaths, const float gr, const int nYears, const float meanP, const float timeStep, const float rrr, float current, float reversion, float jumpProb, const float* brownian, const float* jumpSize, const float* jump, float* result) {

// Get the global index for the matrix
unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

if (idx < noPaths) {
// Simulate a forward path
float value = 0;
float curr = current;

for (int ii = 0; ii < nYears; ii++) {
float jumped = (jump[idx+ii] < jumpProb)? 1.0f : 0.0f;

curr += reversion*(meanP - curr)*timeStep + curr*brownian[idx+ii] +
(exp(jumpSize[idx+ii]) - 1)*curr*jumped;
value += pow(1 + gr,ii)*curr/pow((1 + rrr),ii);
}

result[idx] = value;
}
}