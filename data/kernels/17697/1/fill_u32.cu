#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void fill_u32(unsigned int *y, unsigned int elem, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
y[tid] = elem;
}
}