#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void cross_entropy_forward(unsigned int batch_size, unsigned int nclasses, const float* x, const float* t, float* y) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < batch_size) {
// compute max value of slice
float m = x[tid*nclasses];
for(int i = 1; i < nclasses; ++i) {
m = fmaxf(x[tid*nclasses+i], m);
}
// subtract max
for(int i = 0; i < nclasses; ++i) {
y[tid*nclasses+i] = x[tid*nclasses+i]-m;
}
// sum
float s = 0.0f;
for(int i = 0; i < nclasses; ++i) {
s += expf(y[tid*nclasses+i]);
}
// compute ln(s)
float ln_s = logf(s);
// y = (ln_s - y) * t
for(int i = 0; i < nclasses; ++i) {
y[tid*nclasses+i] = (ln_s - y[tid*nclasses+i]) * t[tid*nclasses+i];
}
}
}