#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void sgd_with_momentum(float* w, const float* dw, float learning_rate, float momentum, float* v, unsigned int len) {
int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid < len) {
v[tid] = momentum * v[tid] + dw[tid];
w[tid] -= learning_rate * v[tid];
}
}