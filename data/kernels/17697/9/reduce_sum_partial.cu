#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void reduce_sum_partial(const float* input, float* output, unsigned int len) {
// from http://www.techdarting.com/2014/06/parallel-reduction-in-cuda.html
// Load a segment of the input vector into shared memory
__shared__ float partialSum[2*256];
int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int t = threadIdx.x;
unsigned int start = 2*blockIdx.x*blockDim.x;

if ((start + t) < len)
{
partialSum[t] = input[start + t];
}
else
{
partialSum[t] = 0.0;
}
if ((start + blockDim.x + t) < len)
{
partialSum[blockDim.x + t] = input[start + blockDim.x + t];
}
else
{
partialSum[blockDim.x + t] = 0.0;
}

// Traverse reduction tree
for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
{
__syncthreads();
if (t < stride)
partialSum[t] += partialSum[t + stride];
}
__syncthreads();

// Write the computed sum of the block to the output vector at correct index
if (t == 0 && (globalThreadId*2) < len)
{
output[blockIdx.x] = partialSum[t];
}
}