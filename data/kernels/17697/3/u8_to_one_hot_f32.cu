#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void u8_to_one_hot_f32(const unsigned char* x, unsigned int nclasses, float* y, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
y[tid*nclasses+x[tid]] = 1.0f;
}
}