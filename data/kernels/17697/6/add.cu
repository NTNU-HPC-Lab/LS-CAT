#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void add(const float* x1, const float* x2, float* y, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
y[tid] = x1[tid] + x2[tid];
}
}