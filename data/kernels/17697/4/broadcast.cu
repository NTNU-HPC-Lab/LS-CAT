#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void broadcast(const float* x, float* y, unsigned int c, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
y[tid] = x[tid % c];
}
}