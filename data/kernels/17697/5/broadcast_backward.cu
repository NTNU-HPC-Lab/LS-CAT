#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void broadcast_backward(float* dx, const float* dy, unsigned int c, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
atomicAdd(&dx[tid % c], dy[tid]);
}
}