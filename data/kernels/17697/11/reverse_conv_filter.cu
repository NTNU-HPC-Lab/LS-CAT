#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void reverse_conv_filter(const float* x, float beta, float* y, unsigned int filter_len, unsigned int len) {
int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid < len) {
if (beta == 0.0f) {
for(int i = 0; i < filter_len; ++i) {
y[tid*filter_len + i] = x[tid*filter_len + ((filter_len-1) - i)];
}
}
else {
for(int i = 0; i < filter_len; ++i) {
y[tid*filter_len + i] = x[tid*filter_len + ((filter_len-1) - i)] + beta * y[tid*filter_len + i];
}
}
}
}