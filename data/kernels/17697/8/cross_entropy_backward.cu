#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void cross_entropy_backward(const float* x, float* dx, const float* t, float* dy, unsigned int len) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
dx[tid] = dy[0] * (x[tid] - t[tid]);
}
}