#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
__global__ void u8_to_f32(const unsigned char* x, float* y, unsigned int len) {
const float scale = 1.0f / 255.0f;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < len) {
y[tid] = scale * x[tid];
}
}