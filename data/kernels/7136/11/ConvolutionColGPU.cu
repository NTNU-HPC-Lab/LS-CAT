#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvolutionColGPU(float *d_Dst,float *d_Src,float *d_Filter,int filterR){
int x =threadIdx.x;
int y =threadIdx.y;

float sum=0;

for (int k = -filterR; k <= filterR; k++) {
int d = y + k;

if (d >= 0 && d < blockDim.y) {
sum += d_Src[d * blockDim.x + x] * d_Filter[filterR - k];
}

d_Dst[y * blockDim.x + x] = sum;
}
}