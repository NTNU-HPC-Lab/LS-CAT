#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram( int * hist_out, unsigned char * img_in, int img_w,int img_h,  int nbr_bin){

int tx=threadIdx.x;
int ty=threadIdx.y;
int bx=blockIdx.x;
int by=blockIdx.y;

__shared__ int smem[256];
smem[threadIdx.x]=0;
__syncthreads();

unsigned int col= tx + blockDim.x * bx;
unsigned int row= ty + blockDim.y * by;


int grid_width = gridDim.x * blockDim.x;
int id = row * grid_width + col;

if(row<img_w && col<img_h)
atomicAdd( &(smem[img_in[id]]) ,1);

__syncthreads();

atomicAdd(&(hist_out[threadIdx.x]),smem[threadIdx.x]);


}