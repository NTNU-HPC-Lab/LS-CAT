#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR){
int k;
float sum=0;
int row=blockDim.y*blockIdx.y+threadIdx.y+filterR;
int col=blockDim.x*blockIdx.x+threadIdx.x+filterR;
int newImageW =imageW+filterR*2;
for (k = -filterR; k <= filterR; k++) {

int d = row+ k;

sum += d_Src[col +newImageW* d] * d_Filter[filterR - k];


}
d_Dst[row * newImageW + col] = sum;
}