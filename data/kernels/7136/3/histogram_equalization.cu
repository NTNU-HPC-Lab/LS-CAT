#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_equalization( int * lut, unsigned char * img_out, unsigned char * img_in, int * hist_in, int img_size, int nbr_bin){

int tx=threadIdx.x;
int ty=threadIdx.y;
int bx=blockIdx.x;
int by=blockIdx.y;

__shared__ int smem[256];
smem[ threadIdx.x ] = lut[ threadIdx.x ];
__syncthreads();

unsigned int col= tx + blockDim.x * bx;
unsigned int row= ty + blockDim.y * by;

int grid_width = gridDim.x * blockDim.x;
int id = row * grid_width + col;

// Get the result image
if(id<img_size){

if(smem[img_in[id]] > 255){
img_out[id] = 255;
}
else{
img_out[id] = (unsigned char)smem[img_in[id]];
}
}

}