#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvolutionRowGPU(float *d_Dst,float *d_Src,float *d_Filter,int filterR){
int x =threadIdx.x;
int y =threadIdx.y;
int k;

float sum=0;
for (k = -filterR; k <= filterR; k++) {
int d = x + k;
if (d >= 0 && d < blockDim.x) {
sum += d_Src[y*blockDim.x+d] * d_Filter[filterR- k];
}

d_Dst[y*blockDim.x+x] = sum;
}

}