#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram( int * hist_out, unsigned char * img_in, int img_w,int img_h,  int nbr_bin){

int tx=threadIdx.x;
int ty=threadIdx.y;
int bx=blockIdx.x;
int by=blockIdx.y;

unsigned int col= tx + blockDim.x * bx;

unsigned int row= ty + blockDim.y * by;

int grid_width = gridDim.x * blockDim.x;
int id = row * grid_width + col;

if(id<nbr_bin)
hist_out[id] = 0;

__syncthreads();

if(row<img_w && col<img_h)
atomicAdd( &(hist_out[img_in[id]]), 1);

}