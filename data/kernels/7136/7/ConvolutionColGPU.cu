#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvolutionColGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR){
int k;
double sum=0;
int row=blockDim.y*blockIdx.y+threadIdx.y;
int col=blockDim.x*blockIdx.x+threadIdx.x;

for (k = -filterR; k <= filterR; k++) {

int d = row+ k;

if (d >= 0 && d < imageW) {
sum += d_Src[col +imageW* d] * d_Filter[filterR - k];
}

d_Dst[row * imageW + col] = sum;
}
}