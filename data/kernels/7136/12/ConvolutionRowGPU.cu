#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR){
int k;
float sum=0;
int row=blockDim.y*blockIdx.y+threadIdx.y;
int col=blockDim.x*blockIdx.x+threadIdx.x;

for (k = -filterR; k <= filterR; k++) {

int d = col+ k;

if (d >= 0 && d < imageW) {
sum += d_Src[row * imageW + d] * d_Filter[filterR - k];
}

d_Dst[row * imageW + col] = sum;
}

}