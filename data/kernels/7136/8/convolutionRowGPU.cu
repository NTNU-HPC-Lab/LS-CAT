#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR){
int k;
float sum=0;
int row=blockDim.y*blockIdx.y+threadIdx.y+filterR;
int col=blockDim.x*blockIdx.x+threadIdx.x+filterR;
int newImageW=imageW+filterR*2;

for (k = -filterR; k <= filterR; k++) {

int d = col+ k;


sum += d_Src[row *newImageW + d] * d_Filter[filterR - k];



}
d_Dst[row *newImageW + col] = sum;
}