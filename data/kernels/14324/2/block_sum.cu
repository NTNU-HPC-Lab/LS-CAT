#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void block_sum(const int *input, int *per_block_results, const size_t n)
{
extern __shared__ int sdata[];

unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

// load input into __shared__ memory
int x = 0;
if(i < n)
{
x = input[i];
}
sdata[threadIdx.x] = x;
__syncthreads();

// contiguous range pattern
for(int offset = blockDim.x / 2;
offset > 0;
offset >>= 1)
{
if(threadIdx.x < offset)
{
// add a partial sum upstream to our own
sdata[threadIdx.x] += sdata[threadIdx.x + offset];
}

// wait until all threads in the block have
// updated their partial sums
__syncthreads();
}

// thread 0 writes the final result
if(threadIdx.x == 0)
{
per_block_results[blockIdx.x] = sdata[0];
}
}