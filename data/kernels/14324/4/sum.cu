#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a, int *b, int *c) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
while (i < N) {
c[i] = a[i] + b[i];
i += gridDim.x * blockDim.x;
}
}