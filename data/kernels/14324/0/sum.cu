#include "hip/hip_runtime.h"
#include "includes.h"
/*

blockIdx: block index
threadIdx: thread index within block
blockDim: threads per block (2)
gridDim: blocks per launch (N/2)

*/



#define N 10


__global__ void sum(int *a, int *b, int *c)
{
int i;
i = blockIdx.x * blockDim.x +
threadIdx.x;
c[i] = a[i] + b[i];
}