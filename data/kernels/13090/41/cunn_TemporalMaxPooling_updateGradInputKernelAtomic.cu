#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
// Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + (
threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + (
threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
float *indices_data = indices + blockIdx.x * output_w * input_n + (
threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

int feat = 0;

if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
// For all features
for (feat = 0; feat < input_n; ++feat) {
atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
}
}
}