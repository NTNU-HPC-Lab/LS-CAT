#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitializeMatrix_kernel( int8_t *matrix, int ldm, int rows, int columns) {

int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

if (i < rows && j < columns) {
int offset = i + j * ldm;

matrix[offset] = 0;
if (i >= rows - 2 && j < 1) {
matrix[offset] = 0x0;
}
if (i < 1 && j >= columns - 2) {
matrix[offset] = 0x0;
}
}
}