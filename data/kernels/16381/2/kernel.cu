#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* d_vec, int n) {

int tid = threadIdx.x;

if(threadIdx.x < n) {
int i = d_vec[tid];
d_vec[tid] = i > 5 ? -i : i;
}
}