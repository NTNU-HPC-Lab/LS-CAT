#include "hip/hip_runtime.h"
#include "includes.h"

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)


__global__ void stencil_1d(int *in, int *out)
{
__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
int lindex = threadIdx.x + RADIUS;

temp[lindex] = in[gindex];
if (threadIdx.x < RADIUS)
{
temp[lindex - RADIUS] = in[gindex - RADIUS];
temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
}

__syncthreads();

int result = 0;
for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
result += temp[lindex + offset];

out[gindex-RADIUS] = result;
}