#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < N) Y[i*INCY] *= X[i*INCX];
}