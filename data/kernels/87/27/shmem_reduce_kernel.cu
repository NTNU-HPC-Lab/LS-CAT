#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
extern __shared__ float sdata[];

int myId = threadIdx.x + blockDim.x * blockIdx.x;
int tid  = threadIdx.x;

// load shared mem from global mem
sdata[tid] = d_in[myId];
__syncthreads();            // make sure entire block is loaded!

// do reduction in shared mem
for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
{
if (tid < s)
{
sdata[tid] += sdata[tid + s];
}
__syncthreads();        // make sure all adds at one stage are done!
}

// only thread 0 writes result for this block back to global mem
if (tid == 0)
{
d_out[blockIdx.x] = sdata[0];
}
}