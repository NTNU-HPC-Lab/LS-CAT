#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int clamp(int value, int bound)
{
if (value < 0) {
return 1;
}
if (value < bound) {
return value;
}
return bound - 1;
}
__device__ int index(int x, int y, int width)
{
return (y * width) + x;
}
__device__ const int FILTER_SIZE = 9; __device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;  __device__ void sort_bubble(float *x, int n_size)
{
for (int i = 0; i < n_size - 1; i++)
{
for(int j = 0; j < n_size - i - 1; j++)
{
if (x[j] > x[j+1])
{
float temp = x[j];
x[j] = x[j+1];
x[j+1] = temp;
}
}
}
}
__global__ void median_filter_2d_sm(unsigned char* input, unsigned char* output, int width, int height)
{
__shared__ int sharedPixels[BLOCKDIM + FILTER_SIZE][BLOCKDIM + FILTER_SIZE];

const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

int xBlockLimit_max = blockDim.x - FILTER_HALFSIZE - 1;
int yBlockLimit_max = blockDim.y - FILTER_HALFSIZE - 1;
int xBlockLimit_min = FILTER_HALFSIZE;
int yBlockLimit_min = FILTER_HALFSIZE;

if (threadIdx.x > xBlockLimit_max && threadIdx.y > yBlockLimit_max) {
int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
}
if (threadIdx.x > xBlockLimit_max && threadIdx.y < yBlockLimit_min) {
int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y] = pixel;
}
if (threadIdx.x < xBlockLimit_min && threadIdx.y > yBlockLimit_max) {
int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
}
if (threadIdx.x < xBlockLimit_min && threadIdx.y < yBlockLimit_min) {
int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x][threadIdx.y] = pixel;
}
if (threadIdx.x < xBlockLimit_min) {
int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x][threadIdx.y + FILTER_HALFSIZE] = pixel;
}
if (threadIdx.x > xBlockLimit_max) {
int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;
}
if (threadIdx.y < yBlockLimit_min) {
int i = index(clamp(x,width), clamp(y - FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y] = pixel;
}
if (threadIdx.y > yBlockLimit_max) {
int i = index(clamp(x,width), clamp(y + FILTER_HALFSIZE,height), width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
}
int i = index(x, y, width);
unsigned int pixel = input[i];
sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;

__syncthreads();

if((x<width) && (y<height))
{
const int color_tid = y * width + x;
float windowMedian[MAX_WINDOW*MAX_WINDOW];
int windowElements = 0;

for (int x_iter = 0; x_iter < FILTER_SIZE; x_iter ++)
{
for (int y_iter = 0; y_iter < FILTER_SIZE; y_iter++)
{
if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height)
{
windowMedian[windowElements++] = sharedPixels[threadIdx.x + x_iter][threadIdx.y + y_iter];
}
}
}
sort_bubble(windowMedian,windowElements);
output[color_tid] = windowMedian[windowElements/2];
}
}