#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int index(int x, int y, int width)
{
return (y * width) + x;
}
__device__ const int FILTER_SIZE = 9; __device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;  __device__ void sort_bubble(float *x, int n_size)
{
for (int i = 0; i < n_size - 1; i++)
{
for(int j = 0; j < n_size - i - 1; j++)
{
if (x[j] > x[j+1])
{
float temp = x[j];
x[j] = x[j+1];
x[j+1] = temp;
}
}
}
}
__global__ void median_filter_2d(unsigned char* input, unsigned char* output, int width, int height)
{
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if((x<width) && (y<height))
{
const int color_tid = index(x,y,width);
float windowMedian[MAX_WINDOW*MAX_WINDOW];
int windowElements = 0;
for (int x_iter = x - FILTER_HALFSIZE; x_iter <= x + FILTER_HALFSIZE; x_iter ++)
{
for (int y_iter = y - FILTER_HALFSIZE; y_iter <= y + FILTER_HALFSIZE; y_iter++)
{
if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height)
{
windowMedian[windowElements++] = input[index(x_iter,y_iter,width)];
}
}
}
sort_bubble(windowMedian,windowElements);
output[color_tid] = windowMedian[windowElements/2];
}
}