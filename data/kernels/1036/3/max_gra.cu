#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__device__ __host__ inline double Max(double x, double y) {
x = fabs(x);
y = fabs(y);
return x > y ? x : y;
}
__global__ void max_gra(double* gra, double* max) {
int index = threadIdx.x;
for (int i = 1; i < blockDim.x; i <<= 1) {
if (index % (i << 1) == i) {
gra[index - i] = Max(gra[index - i], gra[index]);
}
__syncthreads();
}
if (index == 0) {
max[0] = gra[0];
}

}