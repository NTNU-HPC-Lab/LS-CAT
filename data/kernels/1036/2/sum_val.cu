#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__global__ void sum_val(double* val, double* r) {
int index = threadIdx.x;
for (int i = 1; i < blockDim.x; i <<= 1) {
if (index % (i << 1) == i) {
val[index - i] += val[index];
}
__syncthreads();
}
if (index == 0) {
r[0] = val[0];
}
}