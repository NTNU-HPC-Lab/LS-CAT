#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__global__ void create_tuple(double* devx, int* pos_x, int* pos_y, double* value, int N) {
int index = threadIdx.x;
if (index < N) {
pos_x[index] = index;
pos_y[index] = index;
value[index] = 2 * cosf(2 * devx[index]);
}
else if(index == N){
pos_x[index] = N;

}
}