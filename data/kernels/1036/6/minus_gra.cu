#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__global__ void minus_gra(double* gra,int size) {
for (int index = blockIdx.x * blockDim.x + threadIdx.x;
index < size;
index += blockDim.x * gridDim.x)
{
gra[index]=0.0-gra[index];
}
}