#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__device__ double sqr(double x) {
return x * x;
}
__global__ void calculate_gra(double* devx, double* gra,int size) {
for (int index = blockIdx.x * blockDim.x + threadIdx.x;
index < size;
index += blockDim.x * gridDim.x)
{
int pre = index - 1;
if (pre < 0) pre += size;
int next = index + 1;
if (next >= size) next -= size;
gra[index] = devx[pre] * sin(2.0 * devx[index] * devx[pre]) + devx[next] * sin(2.0 * devx[index] * devx[next]);
printf("gra %d %d %d %f %f %f\n", pre, index, next, sqr(devx[index]), devx[pre] * sin(2.0 * devx[index] * devx[pre]), gra[index]);
}
}