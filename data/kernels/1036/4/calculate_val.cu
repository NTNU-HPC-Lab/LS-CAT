#include "hip/hip_runtime.h"
#include "includes.h"
//double* x, * devx, * val, * gra, * r, * graMax;
//double* hes_value;
////int size;
//int* pos_x, * pos_y;
//int* csr;
double* x;
//thrust::pair<int, int> *device_pos;
//typedef double (*fp)(double);
//typedef void (*val_fp)(double*, double*, int);
//typedef void (*valsum_fp)(double*, double*,int);
//typedef void (*gra_fp)(double*, double*, int);
//typedef void (*gramin_fp)(double*, double*,int);
//typedef void (*hes_fp)( double*, thrust::pair<int, int>*, double*, int);
//typedef void (*print_fp)(double*, int);
int numSMs;




__device__ void wait() {
for (int i = 1; i <= 10000000; i++);
}
__device__ double sqr(double x) {
return x * x;
}
__global__ void calculate_val(double* devx, double* val, int size) {
for (int index = blockIdx.x * blockDim.x + threadIdx.x;
index < size;
index += blockDim.x * gridDim.x)
{

int pre = index - 1;
if (pre < 0) pre += size;
int next = index + 1;
if (next >= size) next -= size;
val[index] = sqr(sin(devx[pre] * devx[index])) * sqr(sin(devx[next] * devx[index]));

}

//	wait();
}