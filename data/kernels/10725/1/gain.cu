#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gain(int width, int height, float rGain, float gGain, float bGain, float* input, float* output)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if ((x < width) && (y < height))
{
int index = (y * width + x) * 4;
output[index + 0] = input[index + 0] * rGain;
output[index + 1] = input[index + 1] * gGain;
output[index + 2] = input[index + 2] * bGain;
output[index + 3] = input[index + 3];
}
}