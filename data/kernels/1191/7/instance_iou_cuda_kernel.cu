#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void instance_iou_cuda_kernel( int64_t total_gt_instances, const int64_t* __restrict__ nInstance, int nProposal, const int64_t* __restrict__ proposals_idx, const int64_t* __restrict__ proposals_offset, const int64_t* __restrict__ instance_labels, const int64_t* __restrict__ offset_num_gt_instances, const int64_t* __restrict__ batch, const int64_t* __restrict__ instance_pointnum, float* proposals_iou)
{
for (int proposal_id = blockIdx.x; proposal_id < nProposal; proposal_id += gridDim.x)
{
int start = proposals_offset[proposal_id];
int end = proposals_offset[proposal_id + 1];
int sampleIdx = batch[proposals_idx[start]];
int sampleNInstances = nInstance[sampleIdx];
int instanceOffset = offset_num_gt_instances[sampleIdx];
int proposal_total = end - start;
for (int instance_id = threadIdx.x; instance_id < sampleNInstances;
instance_id += blockDim.x)
{
int instance_total = instance_pointnum[instanceOffset + instance_id];
int intersection = 0;
for (int i = start; i < end; i++)
{
int idx = proposals_idx[i];
if ((int)instance_labels[idx] == instance_id + 1)
{ // 0 is reserved for "no instance"
intersection += 1;
}
}

proposals_iou[instanceOffset + instance_id + proposal_id * total_gt_instances] =
(float)intersection /
((float)(proposal_total + instance_total - intersection) + 1e-5);
}
}
}