#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "instance_iou_cuda_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int64_t total_gt_instances = 1;
const int64_t __restrict__ *nInstance = NULL;
hipMalloc(&nInstance, XSIZE*YSIZE);
int nProposal = 1;
const int64_t __restrict__ *proposals_idx = NULL;
hipMalloc(&proposals_idx, XSIZE*YSIZE);
const int64_t __restrict__ *proposals_offset = NULL;
hipMalloc(&proposals_offset, XSIZE*YSIZE);
const int64_t __restrict__ *instance_labels = NULL;
hipMalloc(&instance_labels, XSIZE*YSIZE);
const int64_t __restrict__ *offset_num_gt_instances = NULL;
hipMalloc(&offset_num_gt_instances, XSIZE*YSIZE);
const int64_t __restrict__ *batch = NULL;
hipMalloc(&batch, XSIZE*YSIZE);
const int64_t __restrict__ *instance_pointnum = NULL;
hipMalloc(&instance_pointnum, XSIZE*YSIZE);
float *proposals_iou = NULL;
hipMalloc(&proposals_iou, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
instance_iou_cuda_kernel<<<gridBlock,threadBlock>>>(total_gt_instances,nInstance,nProposal,proposals_idx,proposals_offset,instance_labels,offset_num_gt_instances,batch,instance_pointnum,proposals_iou);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
instance_iou_cuda_kernel<<<gridBlock,threadBlock>>>(total_gt_instances,nInstance,nProposal,proposals_idx,proposals_offset,instance_labels,offset_num_gt_instances,batch,instance_pointnum,proposals_iou);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
instance_iou_cuda_kernel<<<gridBlock,threadBlock>>>(total_gt_instances,nInstance,nProposal,proposals_idx,proposals_offset,instance_labels,offset_num_gt_instances,batch,instance_pointnum,proposals_iou);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}