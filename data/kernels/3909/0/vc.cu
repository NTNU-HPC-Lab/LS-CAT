#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vc(float *dA, float *dB, int N) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < N) {
dA[id] = dB[id];
}
}