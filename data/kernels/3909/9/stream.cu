#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stream(float *dA, float *dB, float *dC, float alpha, int N) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < N) {
dA[id] = dB[id] + alpha * dC[id];
}
}