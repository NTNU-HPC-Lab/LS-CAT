#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel1(float *dW, float *dWcurr, int N) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < N) {
dWcurr[id] = dW[id];
}
}