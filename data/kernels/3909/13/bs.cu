#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bs(float *drand, float *dput, float *dcall, int n) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n) {
float c1 = 0.319381530f;
float c2 = -0.356563782f;
float c3 = 1.781477937f;
float c4 = -1.821255978f;
float c5 = 1.330274429f;

float zero = 0.0f;
float one = 1.0f;
float two = 2.0f;
float temp4 = 0.2316419f;

float oneBySqrt2pi = 0.398942280f;

float d1, d2;
float phiD1, phiD2;
float sigmaSqrtT;
float KexpMinusRT;

float inRand;

inRand = drand[id];

float S = S_LOWER_LIMIT * inRand + S_UPPER_LIMIT * (1.0f - inRand);
float K = K_LOWER_LIMIT * inRand + K_UPPER_LIMIT * (1.0f - inRand);
float T = T_LOWER_LIMIT * inRand + T_UPPER_LIMIT * (1.0f - inRand);
float R = R_LOWER_LIMIT * inRand + R_UPPER_LIMIT * (1.0f - inRand);
float sigmaVal = SIGMA_LOWER_LIMIT * inRand + SIGMA_UPPER_LIMIT * (1.0f - inRand);

sigmaSqrtT = sigmaVal * (float)sqrt(T);

d1 = ((float)log(S / K) + (R + sigmaVal * sigmaVal / two) * T) / sigmaSqrtT;
d2 = d1 - sigmaSqrtT;

KexpMinusRT = K * (float)exp(-R * T);

// phiD1 = phi(d1)
float X = d1;
float absX = (float)abs(X);
float t = one / (one + temp4 * absX);
float y = one - oneBySqrt2pi * (float)exp(-X * X / two) * t * (c1 + t * (c2 + t * (c3 + t * (c4 + t * c5))));
phiD1 = (X < zero) ? (one - y) : y;
// phiD2 = phi(d2)
X = d2;
absX = abs(X);
t = one / (one + temp4 * absX);
y = one - oneBySqrt2pi * (float)exp(-X * X / two) * t * (c1 + t * (c2 + t * (c3 + t * (c4 + t * c5))));
phiD2 = (X < zero) ? (one - y) : y;

dcall[id] = S * phiD1 - KexpMinusRT * phiD2;

// phiD1 = phi(-d1);
X = -d1;
absX = abs(X);
t = one / (one + temp4 * absX);
y = one - oneBySqrt2pi * (float)exp(-X * X / two) * t * (c1 + t * (c2 + t * (c3 + t * (c4 + t * c5))));
phiD1 = (X < zero) ? (one - y) : y;

// phiD2 = phi(-d2);
X = -d2;
absX = abs(X);
t = one / (one + temp4 * absX);
y = one - oneBySqrt2pi * (float)exp(-X * X / two) * t * (c1 + t * (c2 + t * (c3 + t * (c4 + t * c5))));
phiD2 = (X < zero) ? (one - y) : y;

dput[id] = KexpMinusRT * phiD2 - S * phiD1;
}
}