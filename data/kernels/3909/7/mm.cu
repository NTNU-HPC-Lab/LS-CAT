#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mm(float *dA, float *dB, float *dC, int DIM, int N, int GPUN) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id <= GPUN) {
int i = id / DIM;
int j = id % DIM;
float sum = 0.0f;
for (int k = 0; k < DIM; k++) {
sum += dA[i*DIM+k] * dB[k*DIM+j];
}
dC[id] = sum;
}
}