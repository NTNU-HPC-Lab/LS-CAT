#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mm_tiled(float *dA, float *dB, float *dC, int DIM, int N, int GPUN) {
int it, jt, kt, i, j, k;
__shared__ float sA[32][32], sB[32][32];

// (it, jt) => the first element of a specific tile
it = blockIdx.y * 32;
jt = blockIdx.x * 32;

// (i, j) => specific element
i = it + threadIdx.y;
j = jt + threadIdx.x;

if (i*DIM+j <= GPUN) {
float sum = 0.0f;
// per tile loop
for (kt = 0; kt < DIM; kt += 32) {
// copy to shared memory
sA[threadIdx.y][threadIdx.x] = dA[(it+threadIdx.y)*DIM + kt + threadIdx.x];
sB[threadIdx.y][threadIdx.x] = dB[(kt+threadIdx.y)*DIM + jt + threadIdx.x];
__syncthreads();

// two 32x32 small shared (dB[it + 0:31][kt + 0:31], dC[kt+0:31][jt + 0:31]) at this point
for (k = kt; k < kt+32; k++) {
sum += sA[i-it][k-kt] * sB[k-kt][j-jt];
}

__syncthreads();
}
dC[i*DIM+j] = sum;
}
}