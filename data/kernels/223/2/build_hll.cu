#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void build_hll(int n, unsigned int *in, unsigned int *out) {
int offset = (blockIdx.x * blockDim.x + threadIdx.x);
if (offset < n) {
// Extract the parts
unsigned int val = *(in + offset);
int bucket = val >> HLL_BUCKET_WIDTH;

// Update the maximum position
int pos = val & ((1 << HLL_BUCKET_WIDTH) - 1);

// Wait for all the maximums to be sync'd
atomicMax(&out[bucket], pos);
}
}