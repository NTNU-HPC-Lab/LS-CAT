#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extract_hll(int n, char *in, char *out) {
int offset = (blockIdx.x * blockDim.x + threadIdx.x);
if (offset < n) {
uint64_t *hash = (uint64_t*)(in + (HASH_WIDTH * offset));

// Get the first HLL_PREFIX_BITS to determine the bucket
int bucket = hash[0] >> (64 - HLL_PREFIX_BITS);

// Finds the position of the least significant 1 (0 to 64)
int position = __ffsll(hash[1]);

// Adjust for the limit of the bucket
if (position == 0) {
position = HLL_MAX_SCAN - 1;
} else
position = min(position, HLL_MAX_SCAN) - 1;

// Update the output
unsigned int *outp = ((unsigned int*)out) + offset;
*outp = ((bucket << HLL_BUCKET_WIDTH) | position);
}
}