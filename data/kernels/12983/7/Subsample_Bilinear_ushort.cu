#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_ushort(hipTextureObject_t ushort_tex, unsigned short *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
int y0 = tex2D<unsigned short>(ushort_tex, xi-dx, yi-dy);
int y1 = tex2D<unsigned short>(ushort_tex, xi+dx, yi-dy);
int y2 = tex2D<unsigned short>(ushort_tex, xi-dx, yi+dy);
int y3 = tex2D<unsigned short>(ushort_tex, xi+dx, yi+dy);
dst[yo*dst_pitch+xo] = (unsigned short)((y0+y1+y2+y3+2) >> 2);
}
}