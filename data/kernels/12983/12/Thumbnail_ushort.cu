#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Thumbnail_ushort(hipTextureObject_t ushort_tex, int *histogram, int src_width, int src_height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y < src_height && x < src_width)
{
unsigned short pixel = (tex2D<unsigned short>(ushort_tex, x, y) + 128) >> 8;
atomicAdd(&histogram[pixel], 1);
}
}