#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Overlay_Cuda( int x_position, int y_position, unsigned char* main, int main_linesize, unsigned char* overlay, int overlay_linesize, int overlay_w, int overlay_h, unsigned char* overlay_alpha, int alpha_linesize, int alpha_adj_x, int alpha_adj_y)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x >= overlay_w + x_position ||
y >= overlay_h + y_position ||
x < x_position ||
y < y_position ) {

return;
}

int overlay_x = x - x_position;
int overlay_y = y - y_position;

float alpha = 1.0;
if (alpha_linesize) {
alpha = overlay_alpha[alpha_adj_x * overlay_x  + alpha_adj_y * overlay_y * alpha_linesize] / 255.0f;
}

main[x + y*main_linesize] = alpha * overlay[overlay_x + overlay_y * overlay_linesize] + (1.0f - alpha) * main[x + y*main_linesize];
}