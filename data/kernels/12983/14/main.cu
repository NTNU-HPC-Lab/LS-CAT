#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Overlay_Cuda.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int x_position = 1;
int y_position = 1;
unsigned char *main = NULL;
hipMalloc(&main, XSIZE*YSIZE);
int main_linesize = XSIZE*YSIZE;
unsigned char *overlay = NULL;
hipMalloc(&overlay, XSIZE*YSIZE);
int overlay_linesize = XSIZE*YSIZE;
int overlay_w = 1;
int overlay_h = 1;
unsigned char *overlay_alpha = NULL;
hipMalloc(&overlay_alpha, XSIZE*YSIZE);
int alpha_linesize = XSIZE*YSIZE;
int alpha_adj_x = 2;
int alpha_adj_y = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Overlay_Cuda<<<gridBlock,threadBlock>>>(x_position,y_position,main,main_linesize,overlay,overlay_linesize,overlay_w,overlay_h,overlay_alpha,alpha_linesize,alpha_adj_x,alpha_adj_y);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Overlay_Cuda<<<gridBlock,threadBlock>>>(x_position,y_position,main,main_linesize,overlay,overlay_linesize,overlay_w,overlay_h,overlay_alpha,alpha_linesize,alpha_adj_x,alpha_adj_y);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Overlay_Cuda<<<gridBlock,threadBlock>>>(x_position,y_position,main,main_linesize,overlay,overlay_linesize,overlay_w,overlay_h,overlay_alpha,alpha_linesize,alpha_adj_x,alpha_adj_y);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}