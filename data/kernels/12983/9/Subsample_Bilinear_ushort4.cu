#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_ushort4(hipTextureObject_t ushort4_tex, ushort4 *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
ushort4 c0 = tex2D<ushort4>(ushort4_tex, xi-dx, yi-dy);
ushort4 c1 = tex2D<ushort4>(ushort4_tex, xi+dx, yi-dy);
ushort4 c2 = tex2D<ushort4>(ushort4_tex, xi-dx, yi+dy);
ushort4 c3 = tex2D<ushort4>(ushort4_tex, xi+dx, yi+dy);
int4 res;
res.x =  ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
res.y =  ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
res.z =  ((int)c0.z+(int)c1.z+(int)c2.z+(int)c3.z+2) >> 2;
res.w =  ((int)c0.w+(int)c1.w+(int)c2.w+(int)c3.w+2) >> 2;
dst[yo*dst_pitch+xo] = make_ushort4(
(unsigned short)res.x, (unsigned short)res.y, (unsigned short)res.z, (unsigned short)res.w);
}
}