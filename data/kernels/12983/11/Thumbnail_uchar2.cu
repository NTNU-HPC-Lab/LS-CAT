#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Thumbnail_uchar2(hipTextureObject_t uchar2_tex, int *histogram, int src_width, int src_height)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y < src_height && x < src_width)
{
uchar2 pixel = tex2D<uchar2>(uchar2_tex, x, y);
atomicAdd(&histogram[pixel.x], 1);
atomicAdd(&histogram[256 + pixel.y], 1);
}
}