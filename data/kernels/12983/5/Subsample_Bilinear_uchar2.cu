#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_uchar2(hipTextureObject_t uchar2_tex, uchar2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
int xo = blockIdx.x * blockDim.x + threadIdx.x;
int yo = blockIdx.y * blockDim.y + threadIdx.y;

if (yo < dst_height && xo < dst_width)
{
float hscale = (float)src_width / (float)dst_width;
float vscale = (float)src_height / (float)dst_height;
float xi = (xo + 0.5f) * hscale;
float yi = (yo + 0.5f) * vscale;
// 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
// Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
float dx = wh / (0.5f + wh);
float dy = wv / (0.5f + wv);
uchar2 c0 = tex2D<uchar2>(uchar2_tex, xi-dx, yi-dy);
uchar2 c1 = tex2D<uchar2>(uchar2_tex, xi+dx, yi-dy);
uchar2 c2 = tex2D<uchar2>(uchar2_tex, xi-dx, yi+dy);
uchar2 c3 = tex2D<uchar2>(uchar2_tex, xi+dx, yi+dy);
int2 uv;
uv.x = ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
uv.y = ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
dst[yo*dst_pitch2+xo] = make_uchar2((unsigned char)uv.x, (unsigned char)uv.y);
}
}