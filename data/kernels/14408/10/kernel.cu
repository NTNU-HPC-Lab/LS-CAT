#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int *A, int *B, int *counter, int n) {
int tid = threadIdx.x;

if (tid < n) {
for (int j=0; j<n; j++) {
counter[tid*n+j]++;
A[tid*n+j] = B[tid*n+j];
}
}
}