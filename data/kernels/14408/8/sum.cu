#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a, int *b, int *c, int N) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid<N) {
c[tid] = a[tid] + b[tid];
}
}