#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int blockSum(int *b, int size) {
int sum=0, i;
for (i=0; i<size;++i) {
sum += b[i];
}
return sum;
}
__global__ void computeSumPerBlock(int *a, int N) {
//each block has its own sdata_a shared memory area
extern __shared__ int sdata_a[];
int tmp;

//each thread loads 1 element from global to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i<N) {
sdata_a[tid] = a[i];
// Make sure we load all values of a to shared memory before
//compute the sum of each subblock.
__syncthreads();

// All blocks execute this in parallel. Note each block has its own
//shared memory sdata_a.
if (tid == 0) {
tmp = blockSum(sdata_a,blockDim.x);
a[i] = tmp;
}
}
}