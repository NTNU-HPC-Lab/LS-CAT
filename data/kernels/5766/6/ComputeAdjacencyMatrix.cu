#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeAdjacencyMatrix(float* dOut, int* nn, int n, int k)
{
// Get the column that the current thread is responsible for
auto col = blockIdx.x * blockDim.x + threadIdx.x;

// If id is within bounds
if(col < n)
{
auto nnCol = &nn[col * n];
for(auto i = 0; i < k; ++i)
{
dOut[col * n + nnCol[i]] = dOut[col + n * nnCol[i]] = 1.0f;
}
}
}