#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SortDistances(float* dist, int* idMat, int n, int k)
{
// Get the index of the column that the current thread is responsible for
auto col = blockIdx.x * blockDim.x + threadIdx.x;

// IF col is out of bounds, then do nothing
if(col < n)
{
auto id = &idMat[col * n];
for(auto i = 0; i < n; ++i)
id[i] = i;

auto distCol = &dist[col * n];
// Only care about the first k elements being sorted
for (auto i = 0; i < k; ++i)
{
auto minIndex = i;
for (auto j = i + 1; j < n; ++j)
{
if(distCol[j] < distCol[minIndex])
minIndex = j;
}
auto tmp = distCol[minIndex];
distCol[minIndex] = distCol[i];
distCol[i] = tmp;

auto tmpId = id[minIndex];
id[minIndex] = id[i];
id[i] = tmpId;
}
}
}