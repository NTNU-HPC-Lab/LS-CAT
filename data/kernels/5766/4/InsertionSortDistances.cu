#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InsertionSortDistances(float* dist, int* idMat, int n, int k)
{
// Get the index of the column that the current thread is responsible for
auto col = blockIdx.x * blockDim.x + threadIdx.x;

// IF col is out of bounds, then do nothing
if (col < n)
{
auto id = &idMat[col * n];

id[0] = 0;

auto distCol = &dist[col * n];

// Otherwise, sort column 'col'
auto i = 1;
while(i < n)
{
auto x = distCol[i];
auto currIndex = i;
auto j = i - 1;
while(j >= 0 && distCol[j] > x)
{
distCol[j + 1] = distCol[j];
id[j + 1] = id[j];
--j;
}
distCol[j + 1] = x;
id[j + 1] = currIndex;
++i;
}
}
}