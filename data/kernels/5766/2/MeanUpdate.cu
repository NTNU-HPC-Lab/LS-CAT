#include "hip/hip_runtime.h"
#include "includes.h"
/* Start Header
***************************************************************** /
/*!
\file knn-kernel.cu
\author Koh Wen Lin
\brief
Contains the implementation for kmeans clustering on the gpu.
*/
/* End Header
*******************************************************************/
#define KMEAN_BLOCK_SIZE 32
#define KMEAN_BLOCK_SIZE_1D KMEAN_BLOCK_SIZE * KMEAN_BLOCK_SIZE


__global__ void MeanUpdate(float* dMeanIn, unsigned k, unsigned d, int* count)
{
float ooc = 1.0f / max(1, count[threadIdx.x]);
for(int i = 0; i < d; ++i)
dMeanIn[threadIdx.x * d + i] *= ooc;
}