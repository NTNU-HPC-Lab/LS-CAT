#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeLaplacianInPlace(float* d, int n)
{
// Column to sum
auto x = blockIdx.x * blockDim.x + threadIdx.x;

if(x < n)
{
auto dCol = &d[x * n];

for(auto i = 0; i < n; ++i)
{
if(i != x)
{
dCol[x] += dCol[i];
dCol[i] = -dCol[i];
}
}
}
}