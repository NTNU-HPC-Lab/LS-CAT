#include "hip/hip_runtime.h"
#include "includes.h"
/* Start Header
***************************************************************** /
/*!
\file knn-kernel.cu
\author Koh Wen Lin
\brief
Contains the implementation for kmeans clustering on the gpu.
*/
/* End Header
*******************************************************************/
#define KMEAN_BLOCK_SIZE 32
#define KMEAN_BLOCK_SIZE_1D KMEAN_BLOCK_SIZE * KMEAN_BLOCK_SIZE


__global__ void Mean(float* dIn, unsigned n, unsigned d, int* dGroupIn, float* dMeanIn, unsigned k, int* count)
{
// Each thread block to perform its own summation internally(Reduction), then, each thread block will add its result into global counter and sum
extern __shared__ float sDataSumGroupCount[]; // Dynamic allocated shared memory enough to store block-size amount of data and sum of cluster, group and count.

float* sData = sDataSumGroupCount;
float* sSum = sData + KMEAN_BLOCK_SIZE_1D * d;
int* sGroup = (int*)&sDataSumGroupCount[(k + KMEAN_BLOCK_SIZE_1D) * d];
int* sCount = sGroup + KMEAN_BLOCK_SIZE_1D;

const int tx = threadIdx.x;
int tid = blockIdx.x * blockDim.x + tx;

if(tid >= n)
return;

// Clear shared memory
if(tx < k)
{
for(int i = 0; i < d; ++i)
sSum[tx * d + i] = dMeanIn[tx * d + i];
sCount[tx] = count[tx] = 0.0f;
}

// Each thread perform 1 global load for all its feature and its group index
memcpy(&sData[tx * d], &dIn[tid * d], d * sizeof(float));
sGroup[tx] = dGroupIn[tid];

// Clear old mean
memset(dMeanIn, 0, k * d * sizeof(float));

// Ensure all data relavant to block is loaded
__syncthreads();

int clusterId = sGroup[tx];

for(int i = 0; i < d; ++i)
atomicAdd(&sSum[clusterId * d + i], sData[tx * d + i]);
atomicAdd(&sCount[clusterId], 1);

__syncthreads();

if(tx == 0)
{
for(int i = 0; i < k * d; ++i)
atomicAdd(&dMeanIn[i], sSum[i]);

for(int i = 0; i < k; ++i)
atomicAdd(&count[i], sCount[i]);
}
}