#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeSquareDistance(float* dOut, float* dIn, int n, int d)
{
// Load values that will be reused
__shared__ float blockA[KNN_BLOCK_SIZE][KNN_BLOCK_SIZE];
__shared__ float blockB[KNN_BLOCK_SIZE][KNN_BLOCK_SIZE];

// A is responsible for points indexed between aStart and aEnd
auto aStart = blockIdx.x * blockDim.x;
// B is responsible for points indexed between bStart and bEnd
auto bStart = blockIdx.y * blockDim.y;

auto ax = aStart + threadIdx.x;
auto bx = bStart + threadIdx.y;

auto sqDist = 0.0f;

auto numBlocksVertical = (d - 1) / KNN_BLOCK_SIZE + 1;

// Number of blocks that can be stored along the vertical dimension = gridDim.y
// Therefore this loop runs for each block along the vertical dimension
for(auto i = 0; i < numBlocksVertical; ++i)
{
// The i'th block on the vertical
auto startY = i * KNN_BLOCK_SIZE;
auto currY  = startY + threadIdx.y;

// The first part of the algorithm has each thread responsible
// for loading the values into blockA and blockB
if(startY + threadIdx.y < d)
{
if(ax < n)
blockA[threadIdx.y][threadIdx.x] = dIn[ax * d + currY];
if(bx < n)
blockB[threadIdx.y][threadIdx.x] = dIn[(bStart + threadIdx.x) * d + currY];
}

__syncthreads();

// Since <a,a> = a1 * a1 + a2 * a2 + a3 * a3 + ... + ad * ad
// We can compute the partial sum a1 * a1 + a2 * a2 + a3 * a3 + ... + ak * ak s.t k < d
// Each thread is now responsible for computing the partial sum of their respective element
// If the respective element is out of bounds, this loop can be skipped
if(ax < n && bx < n)
for(auto j = 0; j < KNN_BLOCK_SIZE; ++j)
{
auto diff = blockA[j][threadIdx.x] - blockB[j][threadIdx.y];
sqDist += diff * diff;
}
}

if(ax < n && bx < n)
{
dOut[ax * n + bx] = ax == bx ? INFINITY : sqDist;
}
}