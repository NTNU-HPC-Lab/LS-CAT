#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UniformRandom(double *x, hiprandState *global_state){
int tid =  blockIdx.x;
hiprandState local_state;
local_state = global_state[tid];
x[tid] = (double) hiprand_uniform(&local_state);
global_state[tid] = local_state;
}