#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_array_double(double *a,  double value, size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
a[idx] = value;
idx   += blockDim.x * gridDim.x;
}
}