#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad(float *a, float *b, float *c, float scalar, size_t len)
{
size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
while (idx < len) {
c[idx] = a[idx]+scalar*b[idx];
idx   += blockDim.x * gridDim.x;
}
}