#include "hip/hip_runtime.h"
#include "includes.h"

__global__  void simple_add(float* a, float* b, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < n) {
a[i] = a[i] + b[i];
}
}