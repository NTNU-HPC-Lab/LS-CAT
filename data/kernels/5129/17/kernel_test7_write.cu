#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
unsigned int i;
unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
unsigned int* start_ptr = (unsigned int*) _start_ptr;

if (ptr >= (unsigned int*) end_ptr) {
return;
}


for (i = 0;i < BLOCKSIZE/sizeof(unsigned int); i++){
ptr[i] = start_ptr[i];
}

return;
}