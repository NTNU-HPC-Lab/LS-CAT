#include "hip/hip_runtime.h"
#include "includes.h"
#define num_thread 256
#define num_block 256
__global__ void blending_pairs(float *a,float *b,float *c,float *d,float *wei,int width,int height,int w,float A,float error_lm,float error_mm,int class_num)
{
const int tid=threadIdx.x;
const int bid=blockIdx.x;
const int Idx=num_thread*bid+tid;
float r_LM,r_MM, r_center_LM,r_center_MM;
int row,column;
int i,j;
float sum1,sum2;
float st=0.0;
int judge;
float dis;
//float wei;
float weih,result;
int kk=0;
int rmin,rmax,smin,smax;
for(int kkk=Idx;kkk<width*height;kkk=kkk+num_thread*num_block)
{
result=0;
judge=0;
wei[kkk]=0;
kk=0;
sum1=0,sum2=0;
row=kkk/width;
column=kkk%width;
//if(row==1)
//	wei=0;
r_center_LM =d[kkk]-b[kkk]+error_lm;
r_center_MM=d[kkk]-c[kkk]+1.412*error_mm;
if(column-w/2<=0)
rmin=0;
else
rmin = column-w/2;

if(column+w/2>=width-1)
rmax = width-1;
else
rmax = column+w/2;

if(row-w/2<=0)
smin=0;
else
smin = row-w/2;

if(row+w/2>=height-1)
smax = height-1;
else
smax = row+w/2;
for(i=smin;i<=smax;i++)
{
for(j=rmin;j<=rmax;j++)
{
sum1+=b[i*width+j]*b[i*width+j];
sum2+=b[i*width+j];
}
}
//if(column==30&&row==30)
//	result=0;
st=sqrt(sum1/(w*w)-(sum2/(w*w))*(sum2/(w*w)))/ class_num;
for(i=smin;i<=smax;i++)
{
for(j=rmin;j<=rmax;j++)
{
if(fabs(b[kkk]-b[i*width+j])<st)
{
r_LM=d[i*width+j]-b[i*width+j];
r_MM=d[i*width+j]-c[i*width+j];
if((r_center_LM>0&&r_LM<r_center_LM)||(r_center_LM<0&&r_LM>r_center_LM))
{
if((r_center_MM>0&&r_MM<r_center_MM)||(r_center_MM<0&&r_MM>r_center_MM))
{
r_LM=fabs(r_LM)+0.0001;
r_MM=fabs(r_MM)+0.0001;
if(kkk==i*width+j)
judge=1;
dis=float((row-i)*(row-i)+(column-j)*(column-j));
dis=sqrt(dis)/A+1.0;
weih=1.0/(dis* r_LM*r_MM);
wei[kkk]+=weih;
result+=weih*(c[i*width+j]+b[i*width+j]-d[i*width+j]);
kk++;
}
}
}
}
}
if(kk==0)
{
a[kkk]=abs(b[kkk]+c[kkk]-d[kkk])*1000;
wei[kkk]=1000;

}
else
{
if(judge==0)
{
dis=1.0;
r_LM=fabs(d[kkk]-b[kkk])+0.0001;
r_MM=fabs(d[kkk]-c[kkk])+0.0001;
weih=1.0/(dis* r_LM*r_MM);
result+=weih*(b[kkk]+c[kkk]-d[kkk]);
wei[kkk]+=weih;
}
a[kkk]=result;
//if(a[kkk]<0)
//	a[kkk]=(b[kkk]+c[kkk]-d[kkk]);
}
}

}