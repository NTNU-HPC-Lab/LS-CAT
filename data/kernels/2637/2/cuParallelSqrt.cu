#include "hip/hip_runtime.h"
#include "includes.h"
/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
* The modifications are
*      removed texture memory usage
*      removed split query KNN computation
*      added feature extraction with bilinear interpolation
*
* Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
*/

// Includes


// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/


/**
* Gathers k-th smallest distances for each column of the distance matrix in the top.
*
* @param dist        distance matrix
* @param ind         index matrix
* @param width       width of the distance matrix and of the index matrix
* @param height      height of the distance matrix and of the index matrix
* @param k           number of neighbors to consider
*/


/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
* @param k       number of neighbors to consider
*/


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//

/**
* K nearest neighbor algorithm
* - Initialize CUDA
* - Allocate device memory
* - Copy point sets (reference and query points) from host to device memory
* - Compute the distances + indexes to the k nearest neighbors for each query point
* - Copy distances from device to host memory
*
* @param ref_host      reference points ; pointer to linear matrix
* @param ref_nb        number of reference points ; width of the matrix
* @param query_host    query points ; pointer to linear matrix
* @param query_nb      number of query points ; width of the matrix
* @param dim           dimension of points ; height of the matrices
* @param k             number of neighbor to consider
* @param dist_host     distances to k nearest neighbors ; pointer to linear matrix
* @param dist_host     indexes of the k nearest neighbors ; pointer to linear matrix
*
*/
__global__ void cuParallelSqrt(float *dist, int width, int k){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
dist[yIndex*width + xIndex] = sqrt(dist[yIndex*width + xIndex]);
}