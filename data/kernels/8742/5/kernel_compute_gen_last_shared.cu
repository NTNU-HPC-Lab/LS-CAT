#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_last_shared(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t rows, uint32_t cols) {
__shared__ int shared[3][128 + 2];

int ix = ((blockDim.x - 2) * blockIdx.x + threadIdx.x) & (cols - 1);
int iy = (blockIdx.y + threadIdx.y) & (rows - 1);
int id = iy * cols + ix;

int i = threadIdx.y;
int j = threadIdx.x;

uint8_t mine = matrix_src[id]; // keep cell in register
shared[i][j] = mine;
//shared[i][j] = matrix_src[id];

__syncthreads();

if (i == 1 && j > 0 && j < 129){

uint8_t aliveCells = shared[i + 1][j] +  // lower
shared[i - 1][j] +  // upper
shared[i][j + 1] +  // right
shared[i][j - 1] +  // left
shared[i + 1][j + 1] +
shared[i - 1][j - 1] +  //diagonals
shared[i - 1][j + 1] +
shared[i + 1][j - 1];

matrix_dst[id] = (aliveCells == 3 || (aliveCells == 2 && mine)) ? 1 : 0;
}
}