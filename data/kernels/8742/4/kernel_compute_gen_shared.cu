#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_shared(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols) {
extern __shared__ uint8_t shared[];
uint32_t cell = blockIdx.x*blockDim.x + threadIdx.x;

uint32_t x1 = cell     & cols-1; //% cols;
uint32_t x0 = (x1 - 1) & cols-1; //% cols;
uint32_t x2 = (x1 + 1) & cols-1; //% cols;

uint32_t y1 = cell - x1;
uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

//non aumentiamo il numero di thread durante la copia perche' dovremmo sprecare warp_size thread in quel caso
shared[threadIdx.x + 1]                         = matrix_src[x1 + y0];
shared[threadIdx.x + 1 + blockDim.x + 2]        = matrix_src[cell];
shared[threadIdx.x + 1 + (blockDim.x + 2) * 2]  = matrix_src[x1 + y2];


if (threadIdx.x == 0){
shared[0]                     = matrix_src[x0 + y0];
shared[blockDim.x + 2]        = matrix_src[x0 + y1];
shared[(blockDim.x + 2) * 2]  = matrix_src[x0 + y2];
} else if (threadIdx.x == blockDim.x-1){
shared[blockDim.x + 2 - 1]        = matrix_src[x2 + y0];
shared[(blockDim.x + 2) * 2 - 1]  = matrix_src[x2 + y1];
shared[(blockDim.x + 2) * 3 - 1]  = matrix_src[x2 + y2];
}

__syncthreads();

uint32_t scell = threadIdx.x + 1 + blockDim.x + 2;

uint32_t sx1 = scell;//- blockDim.x - 2;    //& cols-1; //% cols;
uint32_t sx0 = (sx1 - 1); //& cols-1; //% cols;
uint32_t sx2 = (sx1 + 1); //& cols-1; //% cols;

uint32_t sy1 = scell - sx1;
uint32_t sy0 = (sy1 - blockDim.x - 2); // cosi sale di una riga //& dim_m1; //% dim;
uint32_t sy2 = (sy1 + blockDim.x + 2); //Scende di una riga //& dim_m1; //% dim;

uint8_t aliveCells = shared[sx0 + sy0] + shared[sx1 + sy0] + shared[sx2 + sy0] + shared[sx0 + sy1] +
shared[sx2 + sy1] + shared[sx0 + sy2] + shared[sx1 + sy2] + shared[sx2 + sy2];

matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && shared[scell])) ? 1 : 0;
}