#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_multiblocks(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols) {
uint32_t cell = blockIdx.x*blockDim.x + threadIdx.x;
//int row = blockIdx.y * blockDim.y + threadIdx.y;
//int col = blockIdx.x * blockDim.x + threadIdx.x;

uint32_t x1 = cell     & cols-1; //% cols;
uint32_t x0 = (x1 - 1) & cols-1; //% cols;
uint32_t x2 = (x1 + 1) & cols-1; //% cols;

uint32_t y1 = cell - x1;
uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] + matrix_src[x0 + y1] +
matrix_src[x2 + y1] + matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];

matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0;
}