#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_dynamic_positions(float *arr, float t)
{
int threadID = threadIdx.x;
int blockID = blockIdx.x;
int threads_per_block = blockDim.x;
int i = blockID * threads_per_block + threadID;

if (threadID == 0 or threadID == 1 or threadID == 2)
{
arr[i] = arr[i] * t;
}
}