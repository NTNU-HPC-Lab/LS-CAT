#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_dynamic_indices(unsigned int *d_all_ib, unsigned int *d_ib, int flip)
{
if (flip == blockIdx.x)
{
d_ib[threadIdx.x] = d_all_ib[blockDim.x * blockIdx.x + threadIdx.x];
d_ib[3 + threadIdx.x] = 0;
}
}