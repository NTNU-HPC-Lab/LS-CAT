#include "hip/hip_runtime.h"
#include "includes.h"
// Optimized using shared memory and on chip memory
// Compile source: $- nvcc src/TokamakSimulation.cu -o nBody -lglut -lm -lGLU -lGL
// Run Executable: $- ./nBody
//To stop hit "control c" in the window you launched it from.
//Make movies https://gist.github.com/JPEGtheDev/db078e1b066543ce40580060eee9c1bf

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

//atomic mass (u)
#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj" //256 vertices-shape (for array simplicity)
#define N 16*16*16

//***********************
// TODO:
//		Check units velocity calculation mag
//		ಠ_ಠ
//***********************

// Globals
float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__device__ float3 getBodyBodyForce(float4 p0, float4 p1){
float3 f;
float dx = p1.x - p0.x;
float dy = p1.y - p0.y;
float dz = p1.z - p0.z;
float r2 = dx*dx + dy*dy + dz*dz;
float inv_r = 1/sqrt(r2);

float force  = (G*p0.w*p1.w)/(r2);// - (H*p0.w*p1.w)/(r2*r2);

f.x = force*dx*inv_r;
f.y = force*dy*inv_r;
f.z = force*dz*inv_r;

return(f);
}
__global__ void getForces(float4 *g_pos, float3 *force, int offset, int device_ct){
int ii;
float3 force_b2b, forceSum;
float4 posMe;
__shared__ float4 shPos[BLOCK];
int id = threadIdx.x + blockDim.x*blockIdx.x;

forceSum.x = 0.0;
forceSum.y = 0.0;
forceSum.z = 0.0;

posMe.x = g_pos[id+offset].x;
posMe.y = g_pos[id+offset].y;
posMe.z = g_pos[id+offset].z;
posMe.w = g_pos[id+offset].w;

for(int j=0; j < gridDim.x*device_ct; j++)
{
shPos[threadIdx.x] = g_pos[threadIdx.x + blockDim.x*j];
__syncthreads();

#pragma unroll 32
for(int i=0; i < blockDim.x; i++)
{
ii = i + blockDim.x*j;
if(ii != id+offset && ii < N)
{
force_b2b = getBodyBodyForce(posMe, shPos[i]);
forceSum.x += force_b2b.x;
forceSum.y += force_b2b.y;
forceSum.z += force_b2b.z;
}
}
}

if(id <N){
force[id].x = forceSum.x;
force[id].y = forceSum.y;
force[id].z = forceSum.z;
}
}