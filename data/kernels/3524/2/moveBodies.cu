#include "hip/hip_runtime.h"
#include "includes.h"
// Optimized using shared memory and on chip memory
// Compile source: $- nvcc src/TokamakSimulation.cu -o nBody -lglut -lm -lGLU -lGL
// Run Executable: $- ./nBody
//To stop hit "control c" in the window you launched it from.
//Make movies https://gist.github.com/JPEGtheDev/db078e1b066543ce40580060eee9c1bf

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

//atomic mass (u)
#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj" //256 vertices-shape (for array simplicity)
#define N 16*16*16

//***********************
// TODO:
//		Check units velocity calculation mag
//		ಠ_ಠ
//***********************

// Globals
float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__global__ void moveBodies(float4 *g_pos, float4 *d_pos, float3 *vel, float3 * force, int offset){
int id = threadIdx.x + blockDim.x*blockIdx.x;
if(id < N){
vel[id].x += ((force[id].x-DAMP*vel[id].x)/d_pos[id].w)*DT;
vel[id].y += ((force[id].y-DAMP*vel[id].y)/d_pos[id].w)*DT;
vel[id].z += ((force[id].z-DAMP*vel[id].z)/d_pos[id].w)*DT;

d_pos[id].x += vel[id].x*DT;
d_pos[id].y += vel[id].y*DT;
d_pos[id].z += vel[id].z*DT;

g_pos[id+offset].x = d_pos[id].x;
g_pos[id+offset].y = d_pos[id].y;
g_pos[id+offset].z = d_pos[id].z;
}
}