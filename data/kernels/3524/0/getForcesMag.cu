#include "hip/hip_runtime.h"
#include "includes.h"
// Optimized using shared memory and on chip memory
// Compile source: $- nvcc src/TokamakSimulation.cu -o nBody -lglut -lm -lGLU -lGL
// Run Executable: $- ./nBody
//To stop hit "control c" in the window you launched it from.
//Make movies https://gist.github.com/JPEGtheDev/db078e1b066543ce40580060eee9c1bf

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

//atomic mass (u)
#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj" //256 vertices-shape (for array simplicity)
#define N 16*16*16

//***********************
// TODO:
//		Check units velocity calculation mag
//		ಠ_ಠ
//***********************

// Globals
float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__device__ float3 getMagForce(float4 p0, float3 v0, float3 dl_tail, float3 dl_head, float I){
//dl is the section of wire
float3 dB, dl;
dl.x = dl_head.x-dl_tail.x;
dl.y = dl_head.y-dl_tail.y;
dl.z = dl_head.z-dl_tail.z;

float rx = p0.x-dl_tail.x;
float ry = p0.y-dl_tail.y;
float rz = p0.z-dl_tail.z;

float r2 = rx*rx+ry*ry+rz*rz;
float inv_r2 = 1/r2;
float inv_r = 1/sqrtf(r2);
float3 rhat = {rx*inv_r, ry*inv_r, rz*inv_r};

//(dl cross rhat)/r2 = force
//gamma is mu0*I/4Pi which simplifies to Ie-7
float gamma = I;
dB.x = gamma*(dl.y*rhat.z-dl.z*rhat.y)*inv_r2;
dB.y = gamma*(dl.z*rhat.x-dl.x*rhat.z)*inv_r2;
dB.z = gamma*(dl.x*rhat.y-dl.y*rhat.x)*inv_r2;

return (dB);
}
__global__ void getForcesMag(float4 *g_pos, float3 *vel, float3 *force, int offset, float3 *g_reactor){

int id = threadIdx.x + blockDim.x*blockIdx.x;
float3 total_force, B, dB, dl_tail, dl_head, velMe;
float4 posMe;
__shared__ float3 shared_r[BLOCK];

total_force.x = B.x = 0.0;
total_force.y = B.y = 0.0;
total_force.z = B.z = 0.0;

posMe.x = g_pos[id+offset].x;
posMe.y = g_pos[id+offset].y;
posMe.z = g_pos[id+offset].z;
posMe.w = g_pos[id+offset].w;

velMe.x = vel[id].x;
velMe.y = vel[id].y;
velMe.z = vel[id].z;

for(int k=0;k<SHAPE_CT;k++){
shared_r[threadIdx.x] = g_reactor[threadIdx.x + blockDim.x*k];
__syncthreads();

for(int j = 1; j<=SHAPE_SIZE; j++){
dl_tail = shared_r[(j-1)];
dl_head = shared_r[(j%SHAPE_SIZE)];
dB = getMagForce(posMe, velMe, dl_tail, dl_head, 1.0); //current[i] =1

B.x += dB.x;
B.y += dB.y;
B.z += dB.z;
}
}

total_force.x = (velMe.y*B.z-velMe.z*B.y);
total_force.y = (velMe.z*B.x-velMe.x*B.z);
total_force.z = (velMe.x*B.y-velMe.y*B.x);

if(id<N){
force[id].x += total_force.x;
force[id].y += total_force.y;
force[id].z += total_force.z;
}
}