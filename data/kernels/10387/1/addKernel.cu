#include "hip/hip_runtime.h"
#include "includes.h"



/*
Location qualifiers

__global__

Defines a kernel.
Runs on the GPU, called from the CPU.
Executed with <<<dim3>>> arguments.


__device__
Runs on the GPU, called from the GPU.
Can be used for variables too.

__host__

Runs on the CPU, called from the CPU.

Qualifiers can be mixed
Eg __host__ __device__ foo()
Code compiled for both CPU and GPU
useful for testing

*/


__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}