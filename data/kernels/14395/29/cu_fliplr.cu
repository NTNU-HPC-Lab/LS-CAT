#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_fliplr(const float* src, float* dst, const int rows, const int cols, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int c = tid % cols;
int r = tid / cols;
dst[tid] = src[(cols - c - 1) + r * cols];
tid += stride;
}
}