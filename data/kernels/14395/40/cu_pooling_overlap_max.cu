#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_pooling_overlap_max(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int sizex, const int sizey, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = rdst;
int csrc = cdst;
int xend = (csrc + sizex - 1);
int yend = (rsrc + sizey - 1);
loc[tid] = (float)(rsrc * colssrc + csrc);
for(int i = rsrc; i <= yend; ++i){
for(int j = csrc; j <= xend; ++j){
if(src[i * colssrc + j] > dst[tid]){
dst[tid] = src[i * colssrc + j];
loc[tid] = (float)(i * colssrc + j);
}
}
}
tid += stride;
}
}