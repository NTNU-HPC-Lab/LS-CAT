#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_divide(const float* numerator, const float* denominator, float* dst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
if(0 == denominator[tid]) dst[tid] = 0.0;
else dst[tid] = __fdividef(numerator[tid], denominator[tid]);
tid += stride;
}
}