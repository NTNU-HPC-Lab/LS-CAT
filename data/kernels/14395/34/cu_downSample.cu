#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_downSample(const float *src, float* dst, const int y_stride, const int x_stride, const int colssrc, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int colsdst = colssrc / x_stride;
if(colssrc % x_stride > 0) ++colsdst;
while(tid < n){
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = rdst * y_stride;
int csrc = cdst * x_stride;
dst[tid] = src[rsrc * colssrc + csrc];
tid += stride;
}
}