#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_minus(const float *A, float *B, const float c, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
B[tid] = __fsub_rd(A[tid], c);
tid += stride;
}
}