#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_padding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int pad = (cols2 - cols1) / 2;
int c1 = tid % cols1;
int r1 = tid / cols1;
int r2 = r1 + pad;
int c2 = c1 + pad;
dst[r2 * cols2 + c2] = src[tid];
tid += stride;
}
}