#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_interpolation(const float* src, float* dst, const int colssrc, const int colsdst, const int _stride, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int csrc = tid % colssrc;
int rsrc = tid / colssrc;
int rdst = rsrc * _stride;
int cdst = csrc * _stride;
dst[rdst * colsdst + cdst] = src[tid];
tid += stride;
}
}