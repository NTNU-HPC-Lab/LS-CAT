#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_relu(const float* src, float* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
if(src[tid] > 0.0) dst[tid] = src[tid];
else dst[tid] = 0.0;
tid += stride;
}
}