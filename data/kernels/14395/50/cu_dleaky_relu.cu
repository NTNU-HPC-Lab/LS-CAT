#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_dleaky_relu(const float* src, float* dst, int n){
const float leaky_relu_alpha = 100.0;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float p = 0.0;
float n = 0.0;
if(src[tid] > 0.0) p = 1;
if(src[tid] < 0.0) n = 1;
n = fdividef(n, leaky_relu_alpha);
dst[tid] = __fadd_rd(p, n);
tid += stride;
}
}