#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_square(const float *A, float *B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
B[tid] = __fmul_rd(A[tid], A[tid]);
tid += stride;
}
}