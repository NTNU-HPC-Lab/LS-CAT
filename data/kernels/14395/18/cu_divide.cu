#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_divide(float *numerator, float denominator, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
numerator[tid] = __fdividef(numerator[tid], denominator);
tid += stride;
}
}