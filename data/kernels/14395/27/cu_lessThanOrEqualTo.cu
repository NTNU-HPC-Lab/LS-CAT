#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_lessThanOrEqualTo(const float* src, float* dst, const float val, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
if(src[tid] <= val) dst[tid] = 1.0;
else dst[tid] = 0.0;
tid += stride;
}
}