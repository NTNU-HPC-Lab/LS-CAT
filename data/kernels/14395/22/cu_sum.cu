#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_sum(const float* src, float* sum, float *global_mem, const int n){
unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
// load input into __shared__ memory
float x = 0;
if(tid < n){
x = src[tid];
}
global_mem[threadIdx.x] = x;
__syncthreads();
// contiguous range pattern
for(int offset = blockDim.x / 2; offset > 0; offset >>= 1){
if(threadIdx.x < offset){
// add a partial sum upstream to our own
global_mem[threadIdx.x] += global_mem[threadIdx.x + offset];
}
// wait until all threads in the block have
// updated their partial sums
__syncthreads();
}
// thread 0 writes the final result
if(threadIdx.x == 0){
sum[blockIdx.x] = global_mem[0];
}
__syncthreads();
}