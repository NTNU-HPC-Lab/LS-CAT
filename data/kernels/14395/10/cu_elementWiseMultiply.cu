#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_elementWiseMultiply(float *A, const float *B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
A[tid] = __fmul_rd(A[tid], B[tid]);
tid += stride;
}
}