#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_minMaxLoc(const float* src, float* minValue, float* maxValue, int* minLoc, int* maxLoc, float* minValCache, float* maxValCache, int*   minLocCache, int*   maxLocCache, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
//int stride = blockDim.x * gridDim.x;
float val = src[0];
int loc = 0;
if(tid < n){
val = src[tid];
loc = tid;
}
maxValCache[threadIdx.x] = val;
minValCache[threadIdx.x] = val;
maxLocCache[threadIdx.x] = loc;
minLocCache[threadIdx.x] = loc;
__syncthreads();
// contiguous range pattern
for(int offset = blockDim.x / 2; offset > 0; offset >>= 1){
if(threadIdx.x < offset){
// add a partial sum upstream to our own
if(maxValCache[threadIdx.x] >= maxValCache[threadIdx.x + offset]){
;
}else{
maxValCache[threadIdx.x] = maxValCache[threadIdx.x + offset];
maxLocCache[threadIdx.x] = maxLocCache[threadIdx.x + offset];
}
if(minValCache[threadIdx.x] <= minValCache[threadIdx.x + offset]){
;
}else{
minValCache[threadIdx.x] = minValCache[threadIdx.x + offset];
minLocCache[threadIdx.x] = minLocCache[threadIdx.x + offset];
}
}
// wait until all threads in the block have
// updated their partial sums
__syncthreads();
}
// thread 0 writes the final result
if(threadIdx.x == 0){
minValue[blockIdx.x] = minValCache[0];
maxValue[blockIdx.x] = maxValCache[0];
minLoc[blockIdx.x] = minLocCache[0];
maxLoc[blockIdx.x] = maxLocCache[0];
}
}