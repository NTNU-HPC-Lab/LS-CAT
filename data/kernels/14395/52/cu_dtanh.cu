#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_dtanh(const float* src, float* dst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float tmp = __fmul_rd(src[tid], src[tid]);
dst[tid] = __fsub_rd(1.0, tmp);
tid += stride;
}
}