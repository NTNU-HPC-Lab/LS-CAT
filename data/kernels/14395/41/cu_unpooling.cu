#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_unpooling(const float* src, const float* loc, float* dst, const int colsdst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int cdst = (int)(loc[tid]) % colsdst;
int rdst = (int)(loc[tid]) / colsdst;
dst[rdst * colsdst + cdst] = src[tid];
tid += stride;
}
}