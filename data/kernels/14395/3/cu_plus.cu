#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_plus(const float *A, float *B, const float c, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
B[tid] = __fadd_rd(A[tid], c);
tid += stride;
}
}