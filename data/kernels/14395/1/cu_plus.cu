#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_plus(const float *A, const float *B, float *C, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
C[tid] = __fadd_rd(A[tid], B[tid]);
tid += stride;
}
}