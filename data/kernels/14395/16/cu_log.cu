#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_log(const float* src, float* dst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
dst[tid] = __logf(src[tid]);
tid += stride;
}
}