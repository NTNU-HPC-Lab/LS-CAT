#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_dsigmoid(const float* src, float* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float tmp = __expf(src[tid]);
float tmp2 = __fadd_rd(tmp, 1.0);
tmp2 = __fmul_rd(tmp2, tmp2);
dst[tid] = fdividef(tmp, tmp2);
tid += stride;
}
}