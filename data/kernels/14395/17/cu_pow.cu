#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_pow(const float* src, float* dst, const float power, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
dst[tid] = powf(src[tid], power);
tid += stride;
}
}