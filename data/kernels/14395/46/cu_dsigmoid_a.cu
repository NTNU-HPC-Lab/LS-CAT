#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_dsigmoid_a(const float* src, float* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float tmp = __fsub_rd(1.0, src[tid]);
dst[tid] = __fmul_rd(tmp, src[tid]);
tid += stride;
}
}