#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_getRange(const float *src, float* dst, const int xstart, const int xend, const int ystart, const int yend, const int colssrc, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int colsdst = xend - xstart + 1;
while(tid < n){
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = rdst + ystart;
int csrc = cdst + xstart;
dst[tid] = src[rsrc * colssrc + csrc];
tid += stride;
}
}