#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_repmat(const float *a, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int c2 = tid % colsdst;
int r2 = tid / colsdst;
int ra = r2 % rowsa;
int ca = c2 % colsa;
dst[tid] = a[ra * colsa + ca];
tid += stride;
}
}