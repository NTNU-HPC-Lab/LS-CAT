#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_multiply(const float* A, const float* B, float * C, int rowsa, int colsa, int rowsb, int colsb, int rowsc, int colsc){
__shared__ float sA[32][32];   // Tile size of 32x32
__shared__ float sB[32][32];
int Row = blockDim.y*blockIdx.y + threadIdx.y;
int Col = blockDim.x*blockIdx.x + threadIdx.x;
float Cvalue = 0.0;
sA[threadIdx.y][threadIdx.x] = 0.0;
sB[threadIdx.y][threadIdx.x] = 0.0;
for (int k = 0; k < (((colsa - 1)/ 32) + 1); k++){
if ( (Row < rowsa) && (threadIdx.x + (k*32)) < colsa){
sA[threadIdx.y][threadIdx.x] = A[(Row*colsa) + threadIdx.x + (k*32)];
}
else{
sA[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();
if ( Col < colsb && (threadIdx.y + k*32) < rowsb){
sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*32)*colsb + Col];
}
else{
sB[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();

for (int j = 0; j < 32; ++j){
Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
}
__syncthreads();
}
if (Row < rowsc && Col < colsc){
C[Row*colsc + Col] = Cvalue;
}
}