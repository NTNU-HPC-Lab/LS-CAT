#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_tanh(const float* src, float* dst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
dst[tid] = tanhf(src[tid]);
tid += stride;
}
}