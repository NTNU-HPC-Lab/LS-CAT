#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_drelu(const float* src, float* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
if(src[tid] > 0.0) dst[tid] = 1.0;
else dst[tid] = 0.0;
tid += stride;
}
}