#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_leaky_relu(const float* src, float* dst, int n){
const float leaky_relu_alpha = 100.0;
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float p = 0.0;
float n = 0.0;
if(src[tid] > 0.0) p = src[tid];
if(src[tid] < 0.0) n = src[tid];
n = fdividef(n, leaky_relu_alpha);
dst[tid] = __fadd_rd(p, n);
tid += stride;
}
}