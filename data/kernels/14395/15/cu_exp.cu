#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_exp(const float* src, float* dst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
dst[tid] = __expf(src[tid]);
tid += stride;
}
}