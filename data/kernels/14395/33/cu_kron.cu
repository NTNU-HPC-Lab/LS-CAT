#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_kron(const float *a, const float* b, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int colsb = colsdst / colsa;
int rowsb = rowsdst / rowsa;
while(tid < n){
int c2 = tid % colsdst;
int r2 = tid / colsdst;
int rb = r2 % rowsb;
int cb = c2 % colsb;
int ra = r2 / rowsb;
int ca = c2 / colsb;
dst[tid] = a[ra * colsa + ca] * b[rb * colsb + cb];
tid += stride;
}
}