#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_divide(const float* numerator, float* dst, const float denominator, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
if(0 == denominator) dst[tid] = 0.0;
else dst[tid] = __fdividef(numerator[tid], denominator);
tid += stride;
}
}