#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_pooling_max(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int stridex, const int stridey, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = rdst * stridey;
int csrc = cdst * stridex;
int xend = (csrc + stridex - 1) > (colssrc - 1) ? (colssrc - 1) : (csrc + stridex - 1);
int yend = (rsrc + stridey - 1) > (rowssrc - 1) ? (rowssrc - 1) : (rsrc + stridey - 1);
loc[tid] = (float)(rsrc * colssrc + csrc);
for(int i = rsrc; i <= yend; ++i){
for(int j = csrc; j <= xend; ++j){
if(src[i * colssrc + j] > dst[tid]){
dst[tid] = src[i * colssrc + j];
loc[tid] = (float)(i * colssrc + j);
}
}
}
tid += stride;
}
}