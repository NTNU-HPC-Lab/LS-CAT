#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_copyMakeBorder(const float *src, float* dst, const int rowssrc, const int colssrc, const int up, const int down, const int left, const int right, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int colsdst = colssrc + left + right;
while(tid < n){
int csrc = tid % colssrc;
int rsrc = tid / colssrc;
int rdst = up + rsrc;
int cdst = left + csrc;
dst[rdst * colsdst + cdst] = src[tid];
tid += stride;
}
}