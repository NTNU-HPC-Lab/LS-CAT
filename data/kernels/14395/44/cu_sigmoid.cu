#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_sigmoid(const float* src, float* dst, int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
float tmp = __fmul_rd(src[tid], -1.0);
tmp = __expf(tmp);
tmp = __fadd_rd(tmp, 1.0);
dst[tid] = __fdividef(1.0, tmp);
tid += stride;
}
}