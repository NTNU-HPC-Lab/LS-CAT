#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_depadding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int pad = (cols1 - cols2) / 2;
int c2 = tid % cols2;
int r2 = tid / cols2;
int r1 = r2 + pad;
int c1 = c2 + pad;
dst[tid] = src[r1 * cols1 + c1];
tid += stride;
}
}