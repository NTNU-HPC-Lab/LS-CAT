#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_elementWiseMultiply(const float *A, const float B, float *C, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
C[tid] = __fmul_rd(A[tid], B);
tid += stride;
}
}