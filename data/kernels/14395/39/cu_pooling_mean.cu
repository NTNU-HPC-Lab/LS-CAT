#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_pooling_mean(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int stridex, const int stridey, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
int cdst = tid % colsdst;
int rdst = tid / colsdst;
int rsrc = rdst * stridey;
int csrc = cdst * stridex;
int xend = (csrc + stridex - 1) > (colssrc - 1) ? (colssrc - 1) : (csrc + stridex - 1);
int yend = (rsrc + stridey - 1) > (rowssrc - 1) ? (rowssrc - 1) : (rsrc + stridey - 1);
loc[tid] = (float)(rsrc * colssrc + csrc);
for(int i = rsrc; i <= yend; ++i){
for(int j = csrc; j <= xend; ++j){
dst[tid] += __fdividef(src[i * colssrc + j], __fmul_rd(yend - rsrc + 1, xend - csrc + 1));
}
}
tid += stride;
}
}