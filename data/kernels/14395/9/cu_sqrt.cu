#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_sqrt(const float *A, float *B, const int n){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
while(tid < n){
B[tid] = sqrtf(A[tid]);
tid += stride;
}
}