#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "scatterKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const unsigned int *d_In = NULL;
hipMalloc(&d_In, XSIZE*YSIZE);
const unsigned int *d_InPos = NULL;
hipMalloc(&d_InPos, XSIZE*YSIZE);
const unsigned int *d_FalseKeyAddresses = NULL;
hipMalloc(&d_FalseKeyAddresses, XSIZE*YSIZE);
unsigned int *d_Out = NULL;
hipMalloc(&d_Out, XSIZE*YSIZE);
unsigned int *d_OutPos = NULL;
hipMalloc(&d_OutPos, XSIZE*YSIZE);
const unsigned int totalFalses = 1;
size_t size = XSIZE*YSIZE;
unsigned int bitPos = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
scatterKernel<<<gridBlock,threadBlock>>>(d_In,d_InPos,d_FalseKeyAddresses,d_Out,d_OutPos,totalFalses,size,bitPos);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
scatterKernel<<<gridBlock,threadBlock>>>(d_In,d_InPos,d_FalseKeyAddresses,d_Out,d_OutPos,totalFalses,size,bitPos);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
scatterKernel<<<gridBlock,threadBlock>>>(d_In,d_InPos,d_FalseKeyAddresses,d_Out,d_OutPos,totalFalses,size,bitPos);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}