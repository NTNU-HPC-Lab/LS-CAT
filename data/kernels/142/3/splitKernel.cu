#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void splitKernel( const unsigned int *d_In, unsigned int *d_Out, size_t size, unsigned int bitPos, unsigned int *lastValue)
{
int threadsPerBlock = blockDim.x * blockDim.y;

int blockId = blockIdx.x + (blockIdx.y * gridDim.x);

int threadId = threadIdx.x + (threadIdx.y * blockDim.x);

int myId = (blockId * threadsPerBlock) + threadId;

if (myId < size)
{
unsigned int _tmp =
(d_In[myId] >> bitPos) & 0x1;
_tmp =
_tmp ? 0 : 1;

d_Out[myId] =
_tmp;

if ((myId == (size - 1)) &&
(lastValue))
{
*lastValue =
_tmp;
}
}
}