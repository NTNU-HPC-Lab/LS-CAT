#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatterKernel( const unsigned int *d_In, const unsigned int *d_FalseKeyAddresses, unsigned int *d_Out, const unsigned int totalFalses, size_t size, unsigned int bitPos)
{
int threadsPerBlock = blockDim.x * blockDim.y;

int blockId = blockIdx.x + (blockIdx.y * gridDim.x);

int threadId = threadIdx.x + (threadIdx.y * blockDim.x);

int myId = (blockId * threadsPerBlock) + threadId;

if (myId < size)
{
unsigned int _myFalseKeyAddress =
d_FalseKeyAddresses[myId];

// Calculate true sort key address
int _trueSortKeyAddress =
myId - _myFalseKeyAddress + totalFalses;

// True sort key?
unsigned int _trueSortKey =
(d_In[myId] >> bitPos) & 0x1;

int _destinationAddress =
_trueSortKey ?
_trueSortKeyAddress :
_myFalseKeyAddress;

d_Out[_destinationAddress] =
d_In[myId];

}
}