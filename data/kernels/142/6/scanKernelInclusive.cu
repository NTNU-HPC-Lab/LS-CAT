#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scanKernelInclusive(int *c, const int *a, size_t size, size_t offset)
{
int myId =
threadIdx.x;

if (((myId - offset) < size) &&
(myId >= offset))
{
c[myId] = a[myId];

__syncthreads();

size_t _stepsLeft =
size;

unsigned int _neighbor =
1;

while (_stepsLeft)
{
int op1 = c[myId];
int op2 = 0;

if ((myId - offset) >= _neighbor)
{
op2 =
c[myId - _neighbor];
}
else
{
break;
}

__syncthreads();

c[myId] =
op1 + op2;

__syncthreads();

_stepsLeft >>= 1;
_neighbor <<= 1;
}

if (offset > 0)
{
c[myId] +=
c[offset - 1];
}
}
}