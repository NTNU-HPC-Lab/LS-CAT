#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void ImageScalingKernel(float *imgOut, float *imgIn, int width, int height)
{
__shared__ float inData[BLOCK_SIZE];
// Get the index of pixel
const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

// Load data to shared variable
inData[threadIdx.x] = imgIn[index];

if ( index < (width*height) ) {
imgOut[index] = inData[threadIdx.x] / (float)255;
}

__syncthreads();
}