#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void setAllValuesKernel(int* houghSpace, int height, int width, float value)
{
int const index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
if (index < height*width) {
houghSpace[index] = value;
}
__syncthreads();
}