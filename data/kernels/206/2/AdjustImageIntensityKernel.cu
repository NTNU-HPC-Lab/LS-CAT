#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void AdjustImageIntensityKernel(float *imgOut, float *imgIn, int width, int height, float lowin, float lowout, float scale)
{
__shared__ float bufData[BLOCK_SIZE];

// Get the index of pixel
const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

// Load data to shared variable
bufData[threadIdx.x] = imgIn[index];

// Check that it's not out of bounds
if (index < (height*width)) {

// Find the according multiplier
float tempLevel = ( bufData[threadIdx.x] - lowin)*scale + lowout;

// Check that it's within required range
if (tempLevel < 0) {
bufData[threadIdx.x] = 0;
}
else if (tempLevel > 1) {
bufData[threadIdx.x] = 1;
}
else {
bufData[threadIdx.x] = tempLevel;
}

// Write data back
imgOut[index] = bufData[threadIdx.x];
}

// Synchronise threads to have the whole image fully processed for output
__syncthreads();
}