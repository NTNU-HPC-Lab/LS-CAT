#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void one_channel_mul_kernel(float *data_l, float *data_r, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
int one_ch_index = 2 * ((threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = data_l[threadId] * data_r[one_ch_index] - data_l[threadId + 1] * data_r[one_ch_index + 1];
result[threadId + 1] = data_l[threadId] * data_r[one_ch_index + 1] + data_l[threadId + 1] * data_r[one_ch_index];
}