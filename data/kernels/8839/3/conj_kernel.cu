#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conj_kernel(float *data, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = data[threadId];
result[threadId + 1] = -data[threadId + 1];
}