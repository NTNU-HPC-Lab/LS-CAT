#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sqr_norm_kernel(int n, float *out, float *data, float rows, float cols)
{
extern __shared__ float sdata[];
int i = blockDim.x * threadIdx.y + threadIdx.x;
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

sdata[i] = 0;
sdata[i] = data[threadId] * data[threadId] + data[threadId + 1] * data[threadId + 1];
__syncthreads();

for (unsigned int s = (blockDim.x * blockDim.y + 1) / 2, old_s = blockDim.x * blockDim.y; s > 0; s >>= 1) {

if (old_s & 1) s += 1;

if (i < s && i + s < old_s) {
sdata[i] += sdata[i + s];
}
old_s = s;
__syncthreads();
}

if (i == 0) {
atomicAdd(&out[blockId / n], sdata[0] / (rows * cols));
}
}