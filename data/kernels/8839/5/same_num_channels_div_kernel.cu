#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void same_num_channels_div_kernel(float *data_l, float *data_r, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = (data_l[threadId] * data_r[threadId] + data_l[threadId + 1] * data_r[threadId + 1]) /
(data_r[threadId] * data_r[threadId] + data_r[threadId + 1] * data_r[threadId + 1]);
result[threadId + 1] = (data_l[threadId + 1] * data_r[threadId] - data_l[threadId] * data_r[threadId + 1]) /
(data_r[threadId] * data_r[threadId] + data_r[threadId + 1] * data_r[threadId + 1]);
}