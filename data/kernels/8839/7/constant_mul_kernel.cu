#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constant_mul_kernel(float *data_l, float constant, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = data_l[threadId] * constant;
result[threadId + 1] = data_l[threadId + 1] * constant;
}