#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scales_channel_mul_kernel(float *data_l, float *data_r, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
int one_ch_index = 2 * ((threadIdx.y * blockDim.x) + threadIdx.x + blockIdx.x * blockDim.x * blockDim.y);

result[threadId] = data_l[threadId] * data_r[one_ch_index] - data_l[threadId + 1] * data_r[one_ch_index + 1];
result[threadId + 1] = data_l[threadId] * data_r[one_ch_index + 1] + data_l[threadId + 1] * data_r[one_ch_index];
}