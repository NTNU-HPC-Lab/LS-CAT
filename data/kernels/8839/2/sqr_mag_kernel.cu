#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sqr_mag_kernel(float *data, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = data[threadId] * data[threadId] + data[threadId + 1] * data[threadId + 1];
result[threadId + 1] = 0;
}