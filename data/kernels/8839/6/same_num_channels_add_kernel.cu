#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void same_num_channels_add_kernel(float *data_l, float *data_r, float *result)
{
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

result[threadId] = data_l[threadId] + data_r[threadId];
result[threadId + 1] = data_l[threadId + 1] + data_r[threadId + 1];
}