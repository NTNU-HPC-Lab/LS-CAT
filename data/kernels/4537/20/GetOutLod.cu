#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < lod_len) {
out_lod0[index] = in_lod[index] - num_erased[in_lod[index]];
}
}