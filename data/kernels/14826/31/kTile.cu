#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
const int idx = blockIdx.x * blockDim.x + threadIdx.x;
const int numThreads = blockDim.x * gridDim.x;
//    const unsigned int numEls = tgtWidth * tgtHeight;
for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
const uint y = i / tgtWidth;
const uint x = i % tgtWidth;
const uint srcY = y % srcHeight;
const uint srcX = x % srcWidth;
tgt[i] = src[srcY * srcWidth + srcX];
}
}