#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void propose_kernel(int64_t *out, int64_t *proposal, const int64_t *rowptr, const int64_t *col, int64_t numel) {

const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
if (thread_idx < numel) {
if (out[thread_idx] != -1)
return; // Only vist blue nodes.

bool has_unmatched_neighbor = false;

for (int64_t i = rowptr[thread_idx]; i < rowptr[thread_idx + 1]; i++) {
auto v = col[i];

if (out[v] < 0)
has_unmatched_neighbor = true; // Unmatched neighbor found.

if (out[v] == -2) {
proposal[thread_idx] = v; // Propose to first red neighbor.
break;
}
}

if (!has_unmatched_neighbor)
out[thread_idx] = thread_idx;
}
}