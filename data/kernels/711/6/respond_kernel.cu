#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void respond_kernel(int64_t *out, const int64_t *proposal, const int64_t *rowptr, const int64_t *col, int64_t numel) {
const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
if (thread_idx < numel) {
if (out[thread_idx] != -2)
return; // Only vist red nodes.

bool has_unmatched_neighbor = false;

for (int64_t i = rowptr[thread_idx]; i < rowptr[thread_idx + 1]; i++) {
auto v = col[i];

if (out[v] < 0)
has_unmatched_neighbor = true; // Unmatched neighbor found.

if (out[v] == -1 && proposal[v] == thread_idx) {
// Match first blue neighbhor v which proposed to u.
out[thread_idx] = min(thread_idx, v);
out[v] = min(thread_idx, v);
break;
}
}

if (!has_unmatched_neighbor)
out[thread_idx] = thread_idx;
}
}