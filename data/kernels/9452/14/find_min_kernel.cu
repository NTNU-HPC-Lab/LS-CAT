#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void find_min_kernel(float * d_out, const float * d_in)
{
// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
extern __shared__ float sdata[];

const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;
const int threadLId = threadIdx.x;

// load shared mem from global mem
sdata[threadLId] = d_in[threadGId];
__syncthreads();            // make sure entire block is loaded!

// do reduction in shared mem
for (unsigned int blockHalfSize = blockDim.x / 2; blockHalfSize > 0; blockHalfSize >>= 1) {
if (threadLId < blockHalfSize) {
sdata[threadLId] = min(sdata[threadLId], sdata[threadLId + blockHalfSize]);
}
__syncthreads();        // make sure all adds at one stage are done!
}

// only thread 0 writes result for this block back to global mem
if (threadLId == 0)
{
d_out[blockIdx.x] = sdata[0];
}
}