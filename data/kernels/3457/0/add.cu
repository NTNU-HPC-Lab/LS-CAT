#include "hip/hip_runtime.h"
#include "includes.h"


#define N 2560
#define M 512
#define BLOCK_SIZE (N/M)
#define RADIUS 5

__global__ void add(double *a, double *b, double *c, int n){
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx < n){
c[idx] = a[idx] + b[idx];
}
}