#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE  16
#define HEADER_SIZE 122
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
* Structure that represents a BMP image.
*/
typedef struct
{
int   width;
int   height;
float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
* Reads a BMP 24bpp file and returns a BMPImage structure.
* Thanks to https://stackoverflow.com/a/9296467
*/
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
////////////////
// TO-DO #5.2 ////////////////////////////////////////////////
// Implement the GPU version of cpu_applyFilter()           //
//                                                          //
// Does it make sense to have a separate gpu_applyFilter()? //
//////////////////////////////////////////////////////////////
float pixel = 0.0f;

for (int h = 0; h < filter_dim; h++)
{
int offset        = h * stride;
int offset_kernel = h * filter_dim;

for (int w = 0; w < filter_dim; w++)
{
pixel += image[offset + w] * matrix[offset_kernel + w];
}
}

return pixel;
}
__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{
float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };

int index_x = blockIdx.x * blockDim.x + threadIdx.x;
int index_y = blockIdx.y * blockDim.y + threadIdx.y;

__shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];


if (index_x < (width - 2) && index_y < (height - 2))
{
int offset_t = index_y * width + index_x;
int offset   = (index_y + 1) * width + (index_x + 1);
int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

sh_block[offset_shared] = image[offset_t];
__syncthreads();

if((threadIdx.y == BLOCK_SIZE - 1))
{
sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
sh_block[offset_shared + BLOCK_SIZE_SH*2] = image[offset_t + 2*width];
}
__syncthreads();

if((threadIdx.x == BLOCK_SIZE - 1))
{
sh_block[offset_shared + 1] = image[offset_t + 1];
sh_block[offset_shared + 2] = image[offset_t + 2];
}
__syncthreads();

if((threadIdx.y == BLOCK_SIZE - 1) && (threadIdx.x == BLOCK_SIZE - 1))
{
sh_block[offset_shared + BLOCK_SIZE_SH + 1] = image[offset_t + width + 1];
sh_block[offset_shared + BLOCK_SIZE_SH*2 + 1] = image[offset_t + width*2 + 1];
sh_block[offset_shared + BLOCK_SIZE_SH + 2] = image[offset_t + width + 2];
sh_block[offset_shared + BLOCK_SIZE_SH*2 + 2] = image[offset_t + width*2 + 2];
}
__syncthreads();

image_out[offset] = gpu_applyFilter(&sh_block[offset_shared],
BLOCK_SIZE_SH, gaussian, 3);
}
}