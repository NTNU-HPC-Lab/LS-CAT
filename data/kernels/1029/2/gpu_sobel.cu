#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE  16
#define HEADER_SIZE 122
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
* Structure that represents a BMP image.
*/
typedef struct
{
int   width;
int   height;
float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
* Reads a BMP 24bpp file and returns a BMPImage structure.
* Thanks to https://stackoverflow.com/a/9296467
*/
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
////////////////
// TO-DO #5.2 ////////////////////////////////////////////////
// Implement the GPU version of cpu_applyFilter()           //
//                                                          //
// Does it make sense to have a separate gpu_applyFilter()? //
//////////////////////////////////////////////////////////////
float pixel = 0.0f;

for (int h = 0; h < filter_dim; h++)
{
int offset        = h * stride;
int offset_kernel = h * filter_dim;

for (int w = 0; w < filter_dim; w++)
{
pixel += image[offset + w] * matrix[offset_kernel + w];
}
}

return pixel;
}
__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
////////////////
// TO-DO #6.1 /////////////////////////////////////
// Implement the GPU version of the Sobel filter //
///////////////////////////////////////////////////
float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
2.0f,  0.0f, -2.0f,
1.0f,  0.0f, -1.0f };
float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
0.0f,  0.0f,  0.0f,
-1.0f, -2.0f, -1.0f };

const int index_x = blockIdx.x*blockDim.x + threadIdx.x;
const int index_y = blockIdx.y*blockDim.y + threadIdx.y;

__shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];


if (index_x < (width - 2) && index_y < (height - 2))
{
int offset_t = index_y * width + index_x;
int offset   = (index_y + 1) * width + (index_x + 1);
int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

sh_block[offset_shared] = image[offset_t];
__syncthreads();

if((threadIdx.y == BLOCK_SIZE - 1))
{
sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
sh_block[offset_shared + BLOCK_SIZE_SH*2] = image[offset_t + 2*width];
}
__syncthreads();

if((threadIdx.x == BLOCK_SIZE - 1))
{
sh_block[offset_shared + 1] = image[offset_t + 1];
sh_block[offset_shared + 2] = image[offset_t + 2];
}
__syncthreads();

if((threadIdx.y == BLOCK_SIZE - 1) && (threadIdx.x == BLOCK_SIZE - 1))
{
sh_block[offset_shared + BLOCK_SIZE_SH + 1] = image[offset_t + width + 1];
sh_block[offset_shared + BLOCK_SIZE_SH*2 + 1] = image[offset_t + width*2 + 1];
sh_block[offset_shared + BLOCK_SIZE_SH + 2] = image[offset_t + width + 2];
sh_block[offset_shared + BLOCK_SIZE_SH*2 + 2] = image[offset_t + width*2 + 2];
}
__syncthreads();


float gx = gpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_x, 3);
float gy = gpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_y, 3);

// Note: The output can be negative or exceed the max. color value
// of 255. We compensate this afterwards while storing the file.
image_out[offset] = sqrtf(gx * gx + gy * gy);
}
}