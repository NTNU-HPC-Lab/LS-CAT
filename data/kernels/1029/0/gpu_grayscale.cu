#include "hip/hip_runtime.h"
#include "includes.h"


#define BLOCK_SIZE  16
#define HEADER_SIZE 122
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
* Structure that represents a BMP image.
*/
typedef struct
{
int   width;
int   height;
float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
* Reads a BMP 24bpp file and returns a BMPImage structure.
* Thanks to https://stackoverflow.com/a/9296467
*/
__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
////////////////
// TO-DO #4.2 /////////////////////////////////////////////
// Implement the GPU version of the grayscale conversion //
///////////////////////////////////////////////////////////
const int h = blockIdx.y*blockDim.y + threadIdx.y;
const int w = blockIdx.x*blockDim.x + threadIdx.x;

int offset_out = h * width;
int offset = offset_out * 3;

if(h < height && w < width)
{
float *pixel = &image[offset + w * 3];
image_out[offset_out + w] = pixel[0] * 0.0722f + // B
pixel[1] * 0.7152f + // G
pixel[2] * 0.2126f;  // R
}

}