#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2hsl_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, float *img_h, float *img_s, unsigned char *img_l)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
float H, S, L;

float var_r = ( (float)img_r[i]/255 );//Convert RGB to [0,1]
float var_g = ( (float)img_g[i]/255 );
float var_b = ( (float)img_b[i]/255 );
float var_min = (var_r < var_g) ? var_r : var_g;
var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
float var_max = (var_r > var_g) ? var_r : var_g;
var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
float del_max = var_max - var_min;               //Delta RGB value

L = ( var_max + var_min ) / 2;
if ( del_max == 0 )//This is a gray, no chroma...
{
H = 0;
S = 0;
}
else                                    //Chromatic data...
{
if ( L < 0.5 )
S = del_max/(var_max+var_min);
else
S = del_max/(2-var_max-var_min );

float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
if( var_r == var_max ){
H = del_b - del_g;
}
else{
if( var_g == var_max ){
H = (1.0/3.0) + del_r - del_b;
}
else{
H = (2.0/3.0) + del_g - del_r;
}
}

}

if ( H < 0 )
H += 1;
if ( H > 1 )
H -= 1;

img_h[i] = H;
img_s[i] = S;
img_l[i] = (unsigned char)(L*255);
}