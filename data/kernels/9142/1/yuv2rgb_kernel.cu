#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yuv2rgb_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, unsigned char *img_y, unsigned char *img_u, unsigned char *img_v){

int i = threadIdx.x + blockDim.x*blockIdx.x;
unsigned char y, cb, cr;

if(i < img_size){

y  = img_y[i];
cb = img_u[i] - 128;
cr = img_v[i] - 128;

img_r[i] = ( y + 1.402 * cr);
img_g[i] = ( y - 0.344 * cb - 0.714 * cr);
img_b[i] = ( y + 1.772 * cb);

}
}