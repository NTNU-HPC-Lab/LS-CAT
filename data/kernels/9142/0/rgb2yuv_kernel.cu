#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2yuv_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, unsigned char *img_y, unsigned char *img_u, unsigned char *img_v) {

int i = threadIdx.x + blockDim.x * blockIdx.x;

if(i < img_size){
int r, g, b;
r = img_r[i];
g = img_g[i];
b = img_b[i];

img_y[i] = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
img_u[i] = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
img_v[i] = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
}
}