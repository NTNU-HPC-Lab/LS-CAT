#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMulInt(int *a, int b, int *c,int ROW, int COLUMNS){

int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * COLUMNS + ix;

if (ix < ROW && iy < COLUMNS)
{
c[idx] = a[idx] * b ;
}
}