#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *a, int *b, int *c, int ROW, int COLUMNS, int temp)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
int sum = 0;
if( col < COLUMNS && row < ROW)
{
for(int i = 0; i < temp; i++)
{
sum += a[row * temp + i] * b[i * COLUMNS + col];
}
c[row * COLUMNS + col] = sum;
}


}