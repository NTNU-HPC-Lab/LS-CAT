#include "hip/hip_runtime.h"
#include "includes.h"
//ADD TWO MATRICES





__global__ void MatAdd(int *a, int *b, int *c, int ROW, int COLUMNS){

int ix = blockIdx.x * blockDim.x + threadIdx.x;
int iy = blockIdx.y * blockDim.y + threadIdx.y;
int idx = iy * COLUMNS + ix;

if (ix < ROW && iy < COLUMNS)
{
c[idx] = a[idx] + b[idx];
}
}