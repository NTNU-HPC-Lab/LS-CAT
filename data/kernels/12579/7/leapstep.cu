#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leapstep(unsigned long n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double dt) {
const unsigned long serial = blockIdx.x * blockDim.x + threadIdx.x;
if (serial < n){
x[serial] += dt * vx[serial];
y[serial] += dt * vy[serial];
z[serial] += dt * vz[serial];
}
}