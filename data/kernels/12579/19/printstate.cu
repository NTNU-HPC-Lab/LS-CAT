#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printstate(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, int tnow){
const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
if(serial < n){
printf("%d, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %d\n", serial, x[serial], y[serial], z[serial], vx[serial], vy[serial], vz[serial], tnow);
}
}