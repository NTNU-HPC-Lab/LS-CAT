#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accel_3_body(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass, double dt){
/*
*  Three body leapfrog: each particle is in a 3 body system with center mass of galaxy 1 and center mass of galaxy 2
*    Because of SOFTPARAMETER, we dont need to determine if thread is computing against itself
*/
const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numofp1 = NUM_P_BASE * NUM_OF_RING_1 + (NUM_OF_RING_1 - 1) * NUM_OF_RING_1 * INC_NUM_P / 2 + 1;
double ax = 0.0, ay = 0.0, az = 0.0, norm1, norm2;
#ifdef SOFTPARA
double tempsp = (pow(pow(x[0] - x[numofp1], 2) + pow(y[0] - y[numofp1], 2) + pow(z[0] - z[numofp1], 2), 1.5) <= RMIN) ? 0.2 * RMIN : SOFTPARAMETER;
double softparameter = (serial == 0 && serial == numofp1) ? tempsp : SOFTPARAMETER;
#else
double softparameter = 0.0;
#endif
norm1 = pow(softparameter + pow(x[serial] - x[0], 2) + pow(y[serial] - y[0], 2) + pow(z[serial] - z[0], 2), 1.5);
norm2 = pow(softparameter + pow(x[serial] - x[numofp1], 2) + pow(y[serial] - y[numofp1], 2) + pow(z[serial] - z[numofp1], 2), 1.5);
if(serial != 0){
ax += -G * mass[0] * (x[serial] - x[0]) / norm1;
ay += -G * mass[0] * (y[serial] - y[0]) / norm1;
az += -G * mass[0] * (z[serial] - z[0]) / norm1;
}
if(serial != numofp1){
ax += -G * mass[numofp1] * (x[serial] - x[numofp1]) / norm2;
ay += -G * mass[numofp1] * (y[serial] - y[numofp1]) / norm2;
az += -G * mass[numofp1] * (z[serial] - z[numofp1]) / norm2;
}
if(serial < n){
vx[serial] += 0.5 * dt * ax;
vy[serial] += 0.5 * dt * ay;
vz[serial] += 0.5 * dt * az;
}
}