#include "hip/hip_runtime.h"
#include "includes.h"
/*
*  This program is a CUDA C program simulating the N-body system
*    of two galaxies as PHY 241 FINAL PROJECTS
*
*/

/*
*  TODO:
*    1. andromeda
*    2. For accel of center of A, only consider accel from center of B. The same for B.
*    3. When the distance between A and B, the soft parameter changed to 0.2Rmin
*    4. report
*    5. presentation
*
*/



/*
**  Modify the constant parameters if neccessary
**    Constant Section
*/
#define PI 3.14159265
#define BUFFERSIZE 256
#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif
//#define SOFTPARAMETER 0.2 * RMIN
// #define AU 149597870700.0
// #define R (77871.0 * 1000.0 / AU)
// #define G (4.0 * pow(PI, 2))
#define G 0.287915013
#define MASS_1 1000              // Center mass of 1st galaxy
#define MASS_2 1000                // Center mass of 2nd galaxy
#define NUM_OF_RING_1 12         // Number of rings in 1st galaxy
#define NUM_OF_RING_2 12          // Number of rings in 2nd galaxy
// #define RING_BASE_1 (R * 0.2)       // Radius of first ring in 1st galaxy
// #define RING_BASE_2 (R * 0.2)       // Radius of first ring in 2nd galaxy
#define NUM_P_BASE 12             // Number of particles in the first ring
#define INC_NUM_P 3               // increment of number of particles each step
// #define INC_R_RING (0.5 * R)      // increment of radius of rings each step
#define PMASS 1             // mass of each particle
#define V_PARAMTER 1            // Parameter adding to initial velocity to make it elliptic
#define RMIN 1
#define ECCEN 0.5
#define RMAX ((1.0 + ECCEN) * RMIN / (1.0 - ECCEN))
#define RING_BASE_1 (RMIN * 0.2)       // Radius of first ring in 1st galaxy
#define RING_BASE_2 (RMIN * 0.2)       // Radius of first ring in 2nd galaxy
#define INC_R_RING (RMIN * 0.05)      // increment of radius of rings each step
#define SOFTPARAMETER 0.000001
/*
*  Major Function Declarations Section
*
*/


/*
*  Functions Implmenetation Section
*
*/

__global__ void accel_3_body(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass, double dt){
/*
*  Three body leapfrog: each particle is in a 3 body system with center mass of galaxy 1 and center mass of galaxy 2
*    Because of SOFTPARAMETER, we dont need to determine if thread is computing against itself
*/
const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int numofp1 = NUM_P_BASE * NUM_OF_RING_1 + (NUM_OF_RING_1 - 1) * NUM_OF_RING_1 * INC_NUM_P / 2 + 1;
double ax = 0.0, ay = 0.0, az = 0.0, norm1, norm2;
#ifdef SOFTPARA
double tempsp = (pow(pow(x[0] - x[numofp1], 2) + pow(y[0] - y[numofp1], 2) + pow(z[0] - z[numofp1], 2), 1.5) <= 0.25 * RMIN) ? 0.5 * RMIN : SOFTPARAMETER;
double softparameter = (serial == 0 && serial == numofp1) ? tempsp : SOFTPARAMETER;
#else
double softparameter = 0.0;
#endif
norm1 = pow(softparameter + pow(x[serial] - x[0], 2) + pow(y[serial] - y[0], 2) + pow(z[serial] - z[0], 2), 1.5);
norm2 = pow(softparameter + pow(x[serial] - x[numofp1], 2) + pow(y[serial] - y[numofp1], 2) + pow(z[serial] - z[numofp1], 2), 1.5);
if(serial != 0){
ax += -G * mass[0] * (x[serial] - x[0]) / norm1;
ay += -G * mass[0] * (y[serial] - y[0]) / norm1;
az += -G * mass[0] * (z[serial] - z[0]) / norm1;
}
if(serial != numofp1){
ax += -G * mass[numofp1] * (x[serial] - x[numofp1]) / norm2;
ay += -G * mass[numofp1] * (y[serial] - y[numofp1]) / norm2;
az += -G * mass[numofp1] * (z[serial] - z[numofp1]) / norm2;
}
if(serial < n){
vx[serial] += 0.5 * dt * ax;
vy[serial] += 0.5 * dt * ay;
vz[serial] += 0.5 * dt * az;
}
}