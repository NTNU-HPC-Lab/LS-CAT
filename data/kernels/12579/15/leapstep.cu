#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leapstep(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double dt){
const unsigned int serial = blockIdx.x * BLOCKSIZE + threadIdx.x;
if(serial < n){
x[serial] += dt * vx[serial];
y[serial] += dt * vy[serial];
z[serial] += dt * vz[serial];
}
}