#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fitness_kernel(int* chromosome, int* collision) {
/*unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
unsigned int stride = blockDim.x * gridDim.x;*/
unsigned int tid = threadIdx.x;
unsigned int bid = blockIdx.x;
int temp = chromosome[bid];
int d = 0;
extern __shared__ int cache[]; // to use the thread-block shared memory
cache[tid] = 0;
if (tid < bid) {
d = abs(temp - chromosome[tid]);
if ((d == 0) || (d == (bid - tid))) {
cache[tid] = 1;
}
else {
cache[tid] = 0;
}
}

__syncthreads();

//Reduction
unsigned int i = blockDim.x / 2;
while (i >0) {
if (tid < i) {
cache[tid] += cache[tid + i];
}
__syncthreads();
i /= 2;
}

if (tid == 0) {
atomicAdd(collision, cache[0]);
}

/*while (index < n) {
temp = chromosome[index];
index += stride;
}*/
}