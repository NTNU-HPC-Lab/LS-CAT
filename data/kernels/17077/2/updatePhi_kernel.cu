#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updatePhi_kernel(int n, bool* d_flags, float* d_energy, float* d_fatigue, float theta) {
unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
unsigned int stride = blockDim.x * gridDim.x;
while (index < n) {
d_flags[index] = (d_energy[index] - d_fatigue[index]) > theta ? true : false;
index += stride;
}
}