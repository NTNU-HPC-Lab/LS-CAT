#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPU_simple_power_and_interbin_kernel(float2 *d_input_complex, float *d_output_power, int nTimesamples, float norm){
int pos_x = blockIdx.x*blockDim.x + threadIdx.x;
int pos_y = blockIdx.y*nTimesamples;

float2 A;
A.x = 0; A.y = 0;

if( pos_x < nTimesamples ) {
A = d_input_complex[pos_y + pos_x];
d_output_power[pos_y + pos_x] = (A.x*A.x + A.y*A.y)*norm;
}
}