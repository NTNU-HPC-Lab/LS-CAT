#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_rotate_internal_kernel(float* dst, const float* src, float theta, const int nx, const int ny)
{
// this is flawed and should not be production
int   src_size = nx * ny;
float xoff     = (0.5f * nx) - 0.5f;
float yoff     = (0.5f * ny) - 0.5f;

int j0      = blockIdx.x * blockDim.x + threadIdx.x;
int jstride = blockDim.x * gridDim.x;

for(int j = j0; j < ny; j += jstride)
{
for(int i = 0; i < nx; ++i)
{
// indices in 2D
float rx = float(i) - xoff;
float ry = float(j) - yoff;
// transformation
float tx = rx * cosf(theta) + -ry * sinf(theta);
float ty = rx * sinf(theta) + ry * cosf(theta);
// indices in 2D
float x = (tx + xoff);
float y = (ty + yoff);
// index in 1D array
int  rz    = j * nx + i;
auto index = [&](int _x, int _y) { return _y * nx + _x; };
// within bounds
int   x1    = floorf(tx + xoff);
int   y1    = floorf(ty + yoff);
int   x2    = x1 + 1;
int   y2    = y1 + 1;
float fxy1  = 0.0f;
float fxy2  = 0.0f;
int   ixy11 = index(x1, y1);
int   ixy21 = index(x2, y1);
int   ixy12 = index(x1, y2);
int   ixy22 = index(x2, y2);
if(ixy11 >= 0 && ixy11 < src_size)
fxy1 += (x2 - x) * src[ixy11];
if(ixy21 >= 0 && ixy21 < src_size)
fxy1 += (x - x1) * src[ixy21];
if(ixy12 >= 0 && ixy12 < src_size)
fxy2 += (x2 - x) * src[ixy12];
if(ixy22 >= 0 && ixy22 < src_size)
fxy2 += (x - x1) * src[ixy22];
dst[rz] += (y2 - y) * fxy1 + (y - y1) * fxy2;
}
}
}