#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_sirt_pixels_kernel(int p, int nx, int dx, float* recon, const float* data)
{
int d0      = blockIdx.x * blockDim.x + threadIdx.x;
int dstride = blockDim.x * gridDim.x;

for(int d = d0; d < dx; d += dstride)
{
float sum = 0.0f;
for(int i = 0; i < nx; ++i)
sum += recon[d * nx + i];
float upd = data[p * dx + d] - sum;
for(int i = 0; i < nx; ++i)
recon[d * nx + i] += upd;
}
}