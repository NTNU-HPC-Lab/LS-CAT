#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_sum_dist_compute(int dy, int dx, int nx, int ny, const int32_t* ones, uint32_t* sum_dist, int p)
{
int nx0      = blockIdx.x * blockDim.x + threadIdx.x;
int nxstride = blockDim.x * gridDim.x;
int dx0      = blockIdx.y * blockDim.y + threadIdx.y;
int dxstride = blockDim.y * gridDim.y;
int dy0      = blockIdx.z * blockDim.z + threadIdx.z;
int dystride = blockDim.z * gridDim.z;

for(int s = dy0; s < dy; s += dystride)
{
for(int d = dx0; d < dx; d += dxstride)
{
uint32_t*      _sum_dist = sum_dist + (s * nx * ny) + (d * nx);
const int32_t* _ones     = ones + (d * nx);
for(int n = nx0; n < nx; n += nxstride)
{
atomicAdd(&_sum_dist[n], (_ones[n] > 0) ? 1 : 0);
}
}
}
}