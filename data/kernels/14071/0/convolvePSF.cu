#include "hip/hip_runtime.h"
#include "includes.h"
/*
============================================================================
Name        :
Author      : Peter Whidden
Version     :
Copyright   :
Description :
============================================================================
*/




static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/*
* Device kernel that compares the provided PSF distribution to the distribution
* around each pixel in the provided image
*/


__global__ void convolvePSF(int width, int height, int imageCount, short *image, short *results, float *psf, int psfRad, int psfDim)
{
// Find bounds of image
const int x = blockIdx.x*32+threadIdx.x;
const int y = blockIdx.y*32+threadIdx.y;
const int minX = max(x-psfRad, 0);
const int minY = max(y-psfRad, 0);
const int maxX = min(x+psfRad, width);
const int maxY = min(y+psfRad, height);
const int dx = maxX-minX;
const int dy = maxY-minY;
if (dx < 1 || dy < 1) return;
// Read Image
/*__shared__*/ float convArea[13][13]; //convArea[dx][dy];
int xCorrection = x-psfRad < 0 ? 0 : psfDim-dx;
int yCorrection = y-psfRad < 0 ? 0 : psfDim-dy;
float sum = 0.0;
for (int i=0; i<dx; ++i)
{
for (int j=0; j<dy; ++j)
{
float value = float(image[0*width*height+(minX+i)*height+minY+j]);
sum += value;
convArea[i][j] = value;
}
}

float sumDifference = 0.0;
for (int i=0; i<dx; ++i)
{
for (int j=0; j<dy; ++j)
{
sumDifference += abs(convArea[i][j]/sum - psf[(i+xCorrection)*psfDim+j+yCorrection] );
}
}

results[0*width*height+x*height+y] = int(1000.0*sumDifference);//*/convArea[psfRad][psfRad]);

}