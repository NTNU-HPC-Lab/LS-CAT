#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tileMatMul(float* matA, float* matB, float* matC, int aRows, int aCols, int bRows, int bCols, int cRows, int cCols)
{
//define row and column values
int Row = blockIdx.y * TILE_DIM + threadIdx.y;
int Col = blockIdx.x * TILE_DIM + threadIdx.x;

//shared memory arrays
__shared__ float sharedMatA[TILE_DIM][TILE_DIM];
__shared__ float sharedMatB[TILE_DIM][TILE_DIM];

float cResultValue = 0.0;

//calculate tiled matrix multiplication on shared memory
for(int i = 0; i < (aCols-1)/TILE_DIM+1; ++i)
{
if(Row < aRows && i*TILE_DIM+threadIdx.x < aCols)
{
sharedMatA[threadIdx.y][threadIdx.x] = matA[Row*aCols + i*TILE_DIM+threadIdx.x];
}
else
sharedMatA[threadIdx.y][threadIdx.x] = 0.0;

if(Col < bCols && i*TILE_DIM+threadIdx.y < cRows)
sharedMatB[threadIdx.y][threadIdx.x] = matB[(i*TILE_DIM+threadIdx.y)*bCols+Col];
else
sharedMatB[threadIdx.y][threadIdx.x] = 0.0;

__syncthreads();

for(int j = 0; j < TILE_DIM; ++j)
cResultValue += sharedMatA[threadIdx.y][j] * sharedMatB[j][threadIdx.x];

__syncthreads();
}

//put the results in the result matrix
if(Row < cRows && Col < cCols)
matC[Row*cCols+Col] = cResultValue;

}