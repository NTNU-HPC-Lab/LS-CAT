#include "hip/hip_runtime.h"
#include "includes.h"
//Alfred Shaker
//10-13-2015



// CUDA kernel

__global__ void vectorSum(int *a, int *b, int *c, int n)
{
//get the id of global thread
int id = blockIdx.x*blockDim.x+threadIdx.x;

//checks to make sure we're not out of bounds
if(id < n)
c[id] = a[id] + b[id];

}