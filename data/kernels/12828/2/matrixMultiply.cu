#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float* a, float* b, float* c, int n)
{
//use block dimentions to calculate column and row
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

for(int i = 0; i<n; i++)
{
c[row*n + col]+= a[row*n + i] + b[i*n + col];
}
}