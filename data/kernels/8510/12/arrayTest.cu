#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayTest(int n, long *factor, long *arr, long *result, int *const_arr1, long *const_arr2)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i == 0) {
/*
printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
printf("In const %d %d %d\n",const_arr1[0],const_arr1[1],const_arr1[2]);
printf("In const %ld %ld %ld\n",const_arr2[0],const_arr2[1],const_arr2[2]);
*/
}

if (i<n)
{
int idx = i * 3;
result[idx]=arr[idx] * factor[i];
result[idx + 1]=arr[idx + 1] * factor[i];
result[idx + 2]=arr[idx + 2] * factor[i];
/*
printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld %ld %ld] \n", factor[i],
arr[idx],arr[idx+1],arr[idx+2],
result[idx],result[idx+1],result[idx+2]);
*/
}

}