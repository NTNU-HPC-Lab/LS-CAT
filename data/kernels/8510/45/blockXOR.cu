#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blockXOR(int size, const char *input, char *output, long key) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix * 8 < size) {
((long *)output)[ix] = ((const long *)input)[ix] ^ key;
}
}