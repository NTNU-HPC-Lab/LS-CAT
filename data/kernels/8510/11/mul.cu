#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul(int n, long *a, long *b, long *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] * b[i];
printf("CUDA KERNEL MUL %ld * %ld = %ld \n",a[i],b[i],sum[i]);
}

}