#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_self(int *size, int *in, int *out) {
const int ix = threadIdx.x + blockIdx.x * blockDim.x;

if (ix < *size) {
out[ix] = in[ix] * 2;
in[ix] = out[ix];
}
}