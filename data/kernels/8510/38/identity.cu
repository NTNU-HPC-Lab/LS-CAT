#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void identity(int size, long *in, long *out) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < size) {
out[ix] = in[ix];
}
}