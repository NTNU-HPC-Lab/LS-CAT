#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_self(int size, long *inout) {
const int ix = threadIdx.x + blockIdx.x * blockDim.x;

if (ix < size) {
inout[ix] = inout[ix] * 2;
}
}