#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, long *a, long *b, long *sum)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
sum[i] = a[i] + b[i];
printf("CUDA KERNEL ADD %ld + %ld = %ld \n",a[i],b[i],sum[i]);
}

}