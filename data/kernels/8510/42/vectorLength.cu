#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorLength(int size, const double *x, const double *y, double *len) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < size) {
len[ix] = sqrt(x[ix] * x[ix] + y[ix] * y[ix]);
}
}