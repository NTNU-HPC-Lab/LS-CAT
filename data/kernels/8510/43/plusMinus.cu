#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void plusMinus(int size, const double *base, const float *deviation, double *a, float *b) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < size) {
a[ix] = base[ix] - deviation[ix];
b[ix] = base[ix] + deviation[ix];
}
}