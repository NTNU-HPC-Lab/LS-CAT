#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2o(int *size, const long *in, long *out) {
const int ix = threadIdx.x + blockIdx.x * blockDim.x;

if (ix < *size) {
out[ix] = in[ix] * 2;
}
}