#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void identity(int *size, const int *input, int *output) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < *size) {
output[ix] = input[ix];
}
}