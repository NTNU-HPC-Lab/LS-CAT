#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IntDataPointIdentity(int *size, const int *inputX, const int *inputY, int *outputX, int *outputY, int *length) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < *size) {
// copy int array
const int *inArrayBody = &inputX[ix* *length];
int *outArrayBody = &outputX[ix* *length];

for (long i = 0; i < *length; i++) {
outArrayBody[i] = inArrayBody[i];
}

// copy int scalar value
outputY[ix] = inputY[ix];
}
}