#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MNKernel(int count, long * Md, long *Nd, long *Pd, int width) {
// 2D thread ID
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;
// Pvalue stores the Pd element that is computed by the thread
long Pvalue = 0;
for (int k=0; k < width; k++)
Pvalue += Md[row * width + k] * Nd[k * width + col];
Pd[row * width + col] = Pvalue;
}