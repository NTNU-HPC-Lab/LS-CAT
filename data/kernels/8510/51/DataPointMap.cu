#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DataPointMap(int size, const double *inputX, const double *inputY, double *output, const double *inFreeArray, int length) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < size) {
// copy int array
const double *inArrayBody = &inputX[ix* length];
double *outArrayBody = &output[ix* length];

for (long i = 0; i < length; i++) {
outArrayBody[i] = inArrayBody[i] + inFreeArray[i];
}
}
}