#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void intArrayAdd(int *size, const int *input, int *output, const int *inFreeArray, int *length) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < *size) {
// copy int array
const int *inArrayBody = &input[ix* *length];
int *outArrayBody = &output[ix* *length];

for (long i = 0; i < *length; i++) {
outArrayBody[i] = inArrayBody[i] + inFreeArray[i];
}
}
}