#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "calculateFinal.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int n = XSIZE*YSIZE;
int *intermediates0 = NULL;
hipMalloc(&intermediates0, XSIZE*YSIZE);
double *intermediates1 = NULL;
hipMalloc(&intermediates1, XSIZE*YSIZE);
double *intermediates2 = NULL;
hipMalloc(&intermediates2, XSIZE*YSIZE);
int *s0 = NULL;
hipMalloc(&s0, XSIZE*YSIZE);
double *s1 = NULL;
hipMalloc(&s1, XSIZE*YSIZE);
double *s2 = NULL;
hipMalloc(&s2, XSIZE*YSIZE);
int k = 1;
int d = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculateFinal<<<gridBlock,threadBlock>>>(n,intermediates0,intermediates1,intermediates2,s0,s1,s2,k,d);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
calculateFinal<<<gridBlock,threadBlock>>>(n,intermediates0,intermediates1,intermediates2,s0,s1,s2,k,d);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
calculateFinal<<<gridBlock,threadBlock>>>(n,intermediates0,intermediates1,intermediates2,s0,s1,s2,k,d);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}