#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateFinal(int n, int *intermediates0, double *intermediates1, double *intermediates2, int *s0, double *s1, double *s2, int k, int d){

if (blockIdx.x > 0) return;

// Only block is invoked.
// loop for every K
for (int clust = threadIdx.y; clust < k; clust+= blockDim.y){
// loop for every dimension(features)
for (int dim = threadIdx.x; dim < d; dim+= blockDim.x) {

// Calculate  S0
// for counts we don't have dimensions
if (dim == 0) {
//count = 0;
for(int z = clust; z < 450*k; z+=k){
{
s0[clust] += intermediates0[z];
}
}
}

// Calculate S1 and S2
int start = clust * d + dim;
int kd    = k * d;
double *s1end = &intermediates1[450 * kd];
double *s1cur = &intermediates1[start];
double *s2cur = &intermediates2[start];

for (; s1cur < s1end; s1cur += kd, s2cur += kd)
{
s1[start] += *s1cur;
s2[start] += *s2cur;
}
}
}
}