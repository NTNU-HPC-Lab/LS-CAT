#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_l(int *size, long *in, long *out) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;

if (ix < *size) {
out[ix] = in[ix] * 2;
}
}