#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateIntermediates(int n, double *xs, int *cluster_index, int *intermediates0, double *intermediates1, double *intermediates2, int k, int d){


int blocksize = n / 450 + 1;
int start = blockIdx.x * blocksize;
int end1 = start + blocksize;
int end;
if (end1>n) end = n;
else end = end1;

if (end > n ) return;
// loop for every K
for (int clust = threadIdx.y; clust < k; clust+= blockDim.y){
// loop for every dimension(features)
for (int dim = threadIdx.x; dim < d; dim+= blockDim.x) {

// Calculate intermediate S0
// for counts we don't have dimensions
if (dim ==0) {
int count = 0;
for(int z=start; z<end; z++)
{
if(cluster_index[z] == clust) {
count ++;
}
}
intermediates0[blockIdx.x*k+clust] = count;
}

// Calculate intermediate S1 and S2
double sum1 = 0.0;
double sum2 = 0.0;
int idx ;
for (int z=start; z<end; z++) {
if(cluster_index[z] == clust) {
idx = z * d + dim;
sum1 += xs[idx];
sum2 += xs[idx] * xs[idx];

}
}
int index = (blockIdx.x*k*d + clust*d + dim);
intermediates1[index] = sum1;
intermediates2[index] = sum2;
}
}
}