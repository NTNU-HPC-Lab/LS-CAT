#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void intArrayIdentity(int *size, const int *input, int *output, int *length) {
const int ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < *size) {

// copy int array
const int *inArrayBody = &input[ix* *length];

int *outArrayBody = &output[ix* *length];

for (long i = 0; i < *length; i++) {
outArrayBody[i] = inArrayBody[i];
}
}
}