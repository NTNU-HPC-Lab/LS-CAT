#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2(int size, long *in, long *out) {
const int ix = threadIdx.x + blockIdx.x * blockDim.x;

if (ix < size) {
out[ix] = in[ix] * 2;
}
}