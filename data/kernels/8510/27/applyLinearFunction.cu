#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void applyLinearFunction(int *size, const short *x, short *y, short *a, short *b) {
const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
if (ix < *size) {
y[ix] = *a + *b * x[ix];
}
}