#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortKernelSimple(int *arr, int arr_len, int odd)
{
int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + odd;
if (i < arr_len - 1)
{
//Even
int a = arr[i];
int b = arr[i + 1];
if (a > b)
{
arr[i] = b;
arr[i + 1] = a;
}
}
}