#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int greatest_row;  __device__  void swap(float* arr, int ind_a, int ind_b)
{
float tmp = arr[ind_a];
arr[ind_a] = arr[ind_b];
arr[ind_b] = tmp;
}
__global__ void swapRow(float* mat, float* b, int cols, int num_block, int k)
{
int row_i = greatest_row;
if (k != row_i) //If the same row don't swap.
{
int row_k = k*cols;
int swap_row = row_i*cols;
//	Calc. swap interval
int i = threadIdx.x + blockIdx.x * blockDim.x;
// Swap matrix
for (; i < cols; i += num_block*blockDim.x)
swap(mat, swap_row + i, row_k + i);
// Swap b
if(blockIdx.x == 0 && threadIdx.x == 0)
swap(b, row_i, k);
}
}