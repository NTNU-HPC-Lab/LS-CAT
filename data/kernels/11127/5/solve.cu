#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solve(float* mat, float* b, float* x, int rows, int cols)
{
int n = blockIdx.x*threads1D + threadIdx.x;
if (n < rows) //Ensure bounds
x[n] = b[n] / mat[n * cols + n];
}