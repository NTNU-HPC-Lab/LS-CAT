#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortKernelMulti(int *arr, int arr_len, int num_elem, int oddEven)
{
int i = 2 * (blockIdx.x * blockDim.x * num_elem) + oddEven;
int iterEnd = min(arr_len - 1, i + 2 * blockDim.x *num_elem);
// Increment to thread start index:
i += 2 * threadIdx.x;
// Every thread in block (warp) step by num_elem
for (; i < iterEnd; i += 2 * blockDim.x)
{
//Even
int a = arr[i];
int b = arr[i + 1];
if (a > b)
{
arr[i] = b;
arr[i + 1] = a;
}
}
}