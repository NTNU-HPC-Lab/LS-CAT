#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int greatest_row;  __device__  void swap(float* arr, int ind_a, int ind_b)
{
float tmp = arr[ind_a];
arr[ind_a] = arr[ind_b];
arr[ind_b] = tmp;
}
__global__ void swapRow(float* mat, float* b, float* column_k, int rows, int cols, int k)
{
int row_i = greatest_row;
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (k != row_i) //If the same row don't swap.
{
if (i < cols) //Ensure bounds
{
//Swap:
float tmp = mat[k*cols + i];
mat[k*cols + i] = mat[row_i*cols + i];
mat[row_i*cols + i] = tmp;
}
//Swap vector b:
else if (i == cols)
{
float tmp = b[k];
b[k] = b[row_i];
b[row_i] = tmp;
}
}
//Store column k in a separate array: (A[k,k] is updated since the same warp swaps it).
if (i < rows)
column_k[i] = mat[i*cols + k];
}