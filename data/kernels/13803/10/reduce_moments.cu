#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_moments(float *d_arr, float *d_results, int N)
{
__shared__ float sh_array[pThreads];
int n = blockDim.x * blockIdx.x + threadIdx.x;
// sh_array[threadIdx.x] = 0;
if (n < N){
for (int s = blockDim.x / 2; s > 0; s >>= 1){
if ( threadIdx.x < s)
{
sh_array[threadIdx.x] += d_arr[threadIdx.x + s];
}
__syncthreads();
}

if (threadIdx.x ==0){
d_results[blockIdx.x] = sh_array[0];
// printf("%d %f\n", blockIdx.x, d_results[blockIdx.x]);
}
}
}