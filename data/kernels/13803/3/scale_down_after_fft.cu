#include "hip/hip_runtime.h"
#include "includes.h"






__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez, int N_grid, int N_grid_all){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int k = blockIdx.z*blockDim.z + threadIdx.z;
int index = k*N_grid*N_grid + j*N_grid + i;

if(i<N_grid && j<N_grid && k<N_grid){
d_Ex[index] /= float(N_grid_all);
d_Ey[index] /= float(N_grid_all);
d_Ez[index] /= float(N_grid_all);
}
}