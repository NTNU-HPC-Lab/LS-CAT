#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "reduce_fields.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_rho = NULL;
hipMalloc(&d_rho, XSIZE*YSIZE);
float *d_Ex = NULL;
hipMalloc(&d_Ex, XSIZE*YSIZE);
float *d_Ey = NULL;
hipMalloc(&d_Ey, XSIZE*YSIZE);
float *d_Ez = NULL;
hipMalloc(&d_Ez, XSIZE*YSIZE);
float *d_Rrho = NULL;
hipMalloc(&d_Rrho, XSIZE*YSIZE);
float *d_REx = NULL;
hipMalloc(&d_REx, XSIZE*YSIZE);
float *d_REy = NULL;
hipMalloc(&d_REy, XSIZE*YSIZE);
float *d_REz = NULL;
hipMalloc(&d_REz, XSIZE*YSIZE);
int N = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
reduce_fields<<<gridBlock,threadBlock>>>(d_rho,d_Ex,d_Ey,d_Ez,d_Rrho,d_REx,d_REy,d_REz,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
reduce_fields<<<gridBlock,threadBlock>>>(d_rho,d_Ex,d_Ey,d_Ez,d_Rrho,d_REx,d_REy,d_REz,N);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
reduce_fields<<<gridBlock,threadBlock>>>(d_rho,d_Ex,d_Ey,d_Ez,d_Rrho,d_REx,d_REy,d_REz,N);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}