#include "hip/hip_runtime.h"
#include "includes.h"






__global__ void reduce_fields(float *d_rho, float *d_Ex, float* d_Ey, float* d_Ez, float *d_Rrho, float* d_REx, float* d_REy, float* d_REz, int N)
{
__shared__ float rho_array[gThreadsAll];
__shared__ float Ex_array[gThreadsAll];
__shared__ float Ey_array[gThreadsAll];
__shared__ float Ez_array[gThreadsAll];
int n = blockDim.x * blockIdx.x + threadIdx.x;
if (n < N){
for (int s = blockDim.x / 2; s > 0; s >>= 1){
if ( threadIdx.x < s)
{
rho_array[threadIdx.x] += d_rho[threadIdx.x + s];
Ex_array[threadIdx.x] += d_Ex[threadIdx.x + s] * d_Ex[threadIdx.x + s];
Ey_array[threadIdx.x] += d_Ey[threadIdx.x + s] * d_Ey[threadIdx.x + s];
Ez_array[threadIdx.x] += d_Ez[threadIdx.x + s] * d_Ez[threadIdx.x + s];
}
__syncthreads();
}

if (threadIdx.x ==0){
d_Rrho[blockIdx.x] = rho_array[0];
d_REx[blockIdx.x] = Ex_array[0];
d_REy[blockIdx.x] = Ey_array[0];
d_REz[blockIdx.x] = Ez_array[0];
}
}
}