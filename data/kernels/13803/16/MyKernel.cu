#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MyKernel(float* devPtr, size_t pitch, int width, int height)
{
for(int r=0; r<height; ++r){
float* row = (float*)((char*)devPtr + r * pitch);
for (int c = 0; c < width; ++c){
row[c] = 17.3;
}
}
}