#include "hip/hip_runtime.h"
#include "includes.h"






__global__ void set_grid_array_to_value(float *arr, float value, int N_grid){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
int k = blockIdx.z*blockDim.z + threadIdx.z;
int index = k*N_grid*N_grid + j*N_grid + i;

if((i<N_grid) && (j<N_grid) && (k<N_grid)){
arr[index] = value;
}
}