#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addtoall(int* a, int b)
{
int i = threadIdx.x;
atomicAdd(&(a[i]), b);
}