#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSort(float* data,int bucketSize,int* startPoint)
{

//	int L= blockIdx.x * blockDim.x;
int L= blockIdx.x*bucketSize;
int U= L + bucketSize;
int j;
float tmp;
startPoint[blockIdx.x] = L;
for(int i=L+1; i < U; i++)
{
tmp=data[i];
j = i-1;
while(tmp<data[j] && j>=0)
{
data[j+1] = data[j];
j = j-1;
}
data[j+1]=tmp;
}
}