#include "hip/hip_runtime.h"
#include "includes.h"

void printUsage(char* appName);
int parseArgs(char** argv,int* pSize,int *print,int argc);
uint64_t getTime();




__global__ void prime(int *a, int count)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;	//Handle the data at the index
if(tid > count) return;

int can = a[tid];
int counter=3;
//int flag=0;
//float limit = sqrtf((float)can);
float limit = sqrtf(can);
limit = limit+1;
// if even -- get out
if(can%2==0)
{
a[tid] = 1;
}
else
{
for(;counter<=limit;counter+=2)
{
if(can%counter==0)
{
a[tid]=1; // set as prime
break;
}
}
}
}