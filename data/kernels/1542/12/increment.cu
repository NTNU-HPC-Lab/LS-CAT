#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void increment(char* data, size_t length)
{
size_t global_index = threadIdx.x + blockIdx.x * blockDim.x;
if (global_index < length)
data[global_index]++;
}