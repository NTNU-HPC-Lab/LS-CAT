#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyp2p( int4*         __restrict__  dest, int4   const* __restrict__  src, size_t                      num_elems)
{
size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
size_t gridSize = blockDim.x * gridDim.x;

#pragma unroll(5)
for (size_t i=globalId; i < num_elems; i+= gridSize)
{
dest[i] = src[i];
}
}