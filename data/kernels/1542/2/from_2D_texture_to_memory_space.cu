#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void from_2D_texture_to_memory_space(hipTextureObject_t texture_source, float* destination, size_t w, size_t h) {

const uint2 gtid = {
threadIdx.x + blockIdx.x * blockDim.x,
threadIdx.y + blockIdx.y * blockDim.y
};
const auto gtid_serliazed = gtid.x + gtid.y * static_cast<unsigned>(w);

if (gtid.x < w && gtid.y < h) {
const float x = tex2D<float>(texture_source, gtid.x, gtid.y);
printf("Thread %u %u, reading value %4f, and writing to index %3u\n", gtid.x, gtid.y, x, gtid_serliazed);
destination[gtid_serliazed] = x;
}
}