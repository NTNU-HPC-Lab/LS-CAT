#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ind2ptr_kernel(const int64_t *ind_data, int64_t *out_data, int64_t M, int64_t numel) {

int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

if (thread_idx == 0) {
for (int64_t i = 0; i <= ind_data[0]; i++)
out_data[i] = 0;
} else if (thread_idx < numel) {
for (int64_t i = ind_data[thread_idx - 1]; i < ind_data[thread_idx]; i++)
out_data[i + 1] = thread_idx;
} else if (thread_idx == numel) {
for (int64_t i = ind_data[numel - 1] + 1; i < M + 1; i++)
out_data[i] = numel;
}
}