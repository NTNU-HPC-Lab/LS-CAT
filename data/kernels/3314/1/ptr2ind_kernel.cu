#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ptr2ind_kernel(const int64_t *ptr_data, int64_t *out_data, int64_t E, int64_t numel) {

int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

if (thread_idx < numel) {
int64_t idx = ptr_data[thread_idx], next_idx = ptr_data[thread_idx + 1];
for (int64_t i = idx; i < next_idx; i++) {
out_data[i] = thread_idx;
}
}
}