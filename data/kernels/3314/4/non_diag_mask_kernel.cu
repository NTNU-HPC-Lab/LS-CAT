#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void non_diag_mask_kernel(const int64_t *row_data, const int64_t *col_data, bool *out_data, int64_t N, int64_t k, int64_t num_diag, int64_t numel) {

int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

if (thread_idx < numel) {
int64_t r = row_data[thread_idx], c = col_data[thread_idx];

if (k < 0) {
if (r + k < 0) {
out_data[thread_idx] = true;
} else if (r + k >= N) {
out_data[thread_idx + num_diag] = true;
} else if (r + k > c) {
out_data[thread_idx + r + k] = true;
} else if (r + k < c) {
out_data[thread_idx + r + k + 1] = true;
}

} else {
if (r + k >= N) {
out_data[thread_idx + num_diag] = true;
} else if (r + k > c) {
out_data[thread_idx + r] = true;
} else if (r + k < c) {
out_data[thread_idx + r + 1] = true;
}
}
}
}