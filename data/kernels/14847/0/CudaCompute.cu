#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEBUG
//#define HANDLE_ERROR(x) if((x) != 0) cout << "Error!" << endl;

using namespace std;

struct SubBlock{

int * nnz_global_i_idx;
int * nnz_global_o_idx;

int nnz;
int * nnz_local_r_idx;
int * nnz_local_c_idx;
float * nnz_values;
};
//void printSubBlocksInfo(SubBlock * sbs, int nsbs, int mem_b_size);




__global__ void CudaCompute(SubBlock * d_sbs, float * d_x, float * d_y, int nblocks, int mem_b_size, int nrows, int ncols , float * sub_y_arr){
/*
sub_y_arr stores float number, with nblocks rows, mem_b_size columns
*/
//#ifdef DEBUG
//printf("This is Cuda Block # %d: \n", blockIdx.x);
//#endif

//if(blockIdx.x >= nblocks)
//    return;


//SubBlock * work_sb = &d_sbs[blockIdx.x];


//printSubBlocksInfo(work_sb, 1, mem_b_size);

/*
float * x_sub = (float *) malloc(mem_b_size * sizeof(float));
float * y_sub = (float *) malloc(mem_b_size * sizeof(float));
//float * x;


for(int i = 0; i < mem_b_size; i++){
if(work_sb->nnz_global_i_idx[i] > 0 && work_sb->nnz_global_i_idx[i] <= ncols){
// d_x   indexing starts from '1'
// x_sub indexing starts from '0'
x_sub[i] = d_x[work_sb->nnz_global_i_idx[i] - 1];
}
else{
x_sub[i] = 0.0;
}
}

for(int i = 0; i < work_sb->nnz; i++){
int x_sub_idx = work_sb->nnz_local_c_idx[i] - 1;
int y_sub_idx = work_sb->nnz_local_r_idx[i] - 1;
y_sub[y_sub_idx] += work_sb->nnz_values[i] * x_sub[x_sub_idx];
//#ifdef DEBUG
//    printf("This is Cuda Block # %d:  Computing (%d, %d) product as (%f)\n", blockIdx.x, x_sub_idx, y_sub_idx, work_sb->nnz_values[i] * x_sub[x_sub_idx]);
//#endif
}

for(int i = 0; i < mem_b_size; i++){
sub_y_arr[blockIdx.x * mem_b_size + i] = y_sub[i];
}
*/

}