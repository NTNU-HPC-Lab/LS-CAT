#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEBUG
//#define HANDLE_ERROR(x) if((x) != 0) cout << "Error!" << endl;

using namespace std;

struct SubBlock{

int * nnz_global_i_idx;
int * nnz_global_o_idx;

int nnz;
int * nnz_local_r_idx;
int * nnz_local_c_idx;
float * nnz_values;
};
//void printSubBlocksInfo(SubBlock * sbs, int nsbs, int mem_b_size);




__global__ void CudaMergeResults(SubBlock * d_sbs, float * d_x, float * d_y, int nblocks, int mem_b_size, int nrows, int ncols , float * sub_y_arr){
if(blockIdx.x == 0 && threadIdx.x == 0){
for(int i = 0; i < nblocks; i++){
int * outLocs = d_sbs[i].nnz_global_o_idx;
for(int j = 0; j < mem_b_size; j++){

d_y[outLocs[j] - 1] += sub_y_arr[i * mem_b_size + j];
}
}
}
}