#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dev_const(float *px, float k) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
px[tid] = k;
}