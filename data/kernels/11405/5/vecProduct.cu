#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecProduct(int *d_x, int *d_y, int *d_z, int N) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < N) {
d_z[idx] = d_x[idx] * d_y[idx];
}
}