#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *d_x, float *d_y, float *d_z, int Width) {

int idx = threadIdx.x;
int idy = threadIdx.y;

float kernelSum = 0;
if ((idx < Width) && (idy < Width)) {
for (int k = 0; k < Width; ++k) {
kernelSum += d_x[idy * Width + k] * d_y[k * Width + idx];
}
d_z[idy * Width + idx] = kernelSum;
}
}