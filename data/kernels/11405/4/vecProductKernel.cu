#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecProductKernel(float *d_z, const float *d_x, const float *d_y, unsigned int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < N) {
d_z[idx] = d_x[idx] * d_y[idx];
}
}