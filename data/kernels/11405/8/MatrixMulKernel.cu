#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(int *d_x, int *d_y, int *d_z, int Block_Width, int M , int N) {

int row = blockIdx.y*blockDim.y+ threadIdx.y;
int col = blockIdx.x*blockDim.x+ threadIdx.x;

int kernelSum = 0;
if ((row<N) && (col<N)) {
for (int i = 0; i < Block_Width ; ++i) {
kernelSum+=d_x[col * Block_Width + i] * d_y[i * Block_Width + row];
}
}
d_z[row * Block_Width +col] = kernelSum;
}