#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

/********************************************************************
*
* Compute C = A x B
*   where A is a (m x k) matrix
*   where B is a (k x n) matrix
*   where C is a (m x n) matrix
*
* Use shared memory for tiling
*
********************************************************************/

// INSERT KERNEL CODE HERE
unsigned int TiRow = threadIdx.y;
unsigned int TiCol = threadIdx.x;
unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

__shared__ float As[TILE_SZ][TILE_SZ];
__shared__ float Bs[TILE_SZ][TILE_SZ];

float sum = 0;


for(unsigned int TiNum = 0; TiNum < (k-1)/TILE_SZ+1; TiNum++){
if((row < m) && (TiNum * TILE_SZ + TiCol) < k)
As[TiRow][TiCol]= A[row * k + TiNum * TILE_SZ + TiCol];
else
As[TiRow][TiCol] = 0;

if((TiNum * TILE_SZ + TiRow) < k && col < n)
Bs[TiRow][TiCol] = B[(TiNum * TILE_SZ + TiRow) * n + col];
else
Bs[TiRow][TiCol] = 0;
__syncthreads();

//Calculate inner product for the tile
//Checking for matrix size to lower power and practice green computing
if(row < m && col < n)
for(unsigned int TiElem = 0; TiElem < TILE_SZ; TiElem++)
sum = sum + As[TiRow][TiElem]*Bs[TiElem][TiCol];
__syncthreads();

}

//Prevent writing of output to an undefined block
if (row < m && col < n)
C[row * n + col] = sum;
}