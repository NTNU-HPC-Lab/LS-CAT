#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins){

extern __shared__ unsigned int bins_s[];

//Shared Memory
int thid = threadIdx.x;
while(thid < num_bins){

bins_s[thid] = 0u;
thid += blockDim.x;
}
__syncthreads();


//Histogram calculation
unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;

while(element < num_elements){

atomicAdd(&(bins_s[input[element]]), 1);
element += blockDim.x * gridDim.x;
}
__syncthreads();

//Global Memory
thid = threadIdx.x;
while(thid < num_bins){

atomicAdd(&(bins[thid]), bins_s[thid]);
thid += blockDim.x;
}
}