#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

/********************************************************************
*
* Compute C = A x B
*   where A is a (m x k) matrix
*   where B is a (k x n) matrix
*   where C is a (m x n) matrix
*
********************************************************************/

// INSERT KERNEL CODE HERE
int row, col;

row = blockIdx.y*blockDim.y+threadIdx.y;

col = blockIdx.x*blockDim.x+threadIdx.x;


if(( row < m) && (col < n))
{
float acc = 0;

for(int index = 0; index < k; index++)
{
acc = acc + A[row * k + index] * B[index * n + col];
}

C[row * n + col] = acc;

}
}