#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8, unsigned int num_bins) {

// INSERT CODE HERE
int thid = blockIdx.x * blockDim.x + threadIdx.x;

while (thid < num_bins){

//Use local  register value (avoids copying from global twice)
unsigned int reg_bin = bins32[thid];

if(reg_bin > 255){
bins8[thid] = 255u;
}

else{
bins8[thid] = (uint8_t) reg_bin;
}
thid += blockDim.x * gridDim.x;
}

}