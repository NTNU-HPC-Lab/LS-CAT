#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_kernel_optimized(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

// INSERT CODE HERE
extern __shared__ unsigned int bins_s[];

//Shared memory
int thid = threadIdx.x;
while ( thid < num_bins){

bins_s[thid] = 0u;
thid += blockDim.x;
}
__syncthreads();

//Histogram calculation
unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int accumulator = 0;
unsigned int prev_index = 0;

while(element < num_elements){

unsigned int curr_index = input[element];

if(curr_index != prev_index){

atomicAdd(&(bins_s[prev_index]), accumulator);
accumulator = 1;
prev_index = curr_index;

}

else{
accumulator++;
}
element += blockDim.x * gridDim.x;
}
if(accumulator > 0){
atomicAdd(&(bins_s[prev_index]), accumulator);
}
__syncthreads();

//Global memory
thid = threadIdx.x;
while(thid < num_bins){

atomicAdd(&(bins[thid]), bins_s[thid]);
thid += blockDim.x;
}

}