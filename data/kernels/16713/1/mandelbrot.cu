#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float function_a_appli(float x);



__global__ void mandelbrot (  int nb_ligne, int nb_col, float seuil, float x_min, float x_max, float y_min, float y_max, float* res) {
int max_ITER=10000;
int iter=0;
int index_col=threadIdx.x+blockDim.x*blockIdx.x;
int index_ligne=threadIdx.y+blockDim.y*blockIdx.y;
int global_index;
float x,y,xtemp,x0,y0;
if ((index_col >= nb_col) || (index_ligne>=nb_ligne) ) return;
global_index=index_ligne*nb_col+index_col;
x0=((float)index_col/(float)nb_col)*(x_max-x_min)+x_min;
y0=((float)(nb_ligne-index_ligne)/(float)nb_ligne)*(y_max-y_min)+y_min;
x=0;y=0;
while((x*x+y*y <= seuil) && (iter < max_ITER))
{   xtemp = x*x-y*y+x0;
y = 2*x*y+y0;
x = xtemp;
iter++;
}
res[global_index]=((float) iter/(float)max_ITER);
}