#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void counthits(int n, uint *hitsp, unsigned decalage_index) {
hiprandStatePhilox4_32_10_t state;
int index = threadIdx.x + blockIdx.x * blockDim.x;
unsigned hits = 0;
int tries=0;
float x1 ;
float y1 ;
float x2 ;
float y2 ;
// initialise avec un 'seed' egale a zero, choix de la suite pseudo aléatoire numero index+decalage_index (le decalage index permet de changer de suite si on relance...), et commence cette suite à partir de 0
/* Dans la doc Nvidia :
hiprand_init (
unsigned long long seed, unsigned long long sequence,
unsigned long long offset, hiprandState_t *state)*/
hiprand_init(0, index+decalage_index, 0, &state);
float4 rand_vec;
while (tries < n) {
rand_vec=hiprand_uniform4 (&state);
x1 = 2*rand_vec.x-1;
y1 = 2*rand_vec.y-1;
x2 = 2*rand_vec.z-1;
y2 = 2*rand_vec.w-1;
if ( (x1*x1 + y1*y1) < 1) {
hits++;
}
if ( (x2*x2 + y2*y2) < 1 ) {
hits++;
}
tries+=2;
}
hitsp[index]=hits;
}