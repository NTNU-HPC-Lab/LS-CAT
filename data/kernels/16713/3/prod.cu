#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void prod( int taille, float * a, float  b, float *c  ){

int index=threadIdx.x+blockDim.x*blockIdx.x;
if(index>=taille) return;
c[index]=a[index]*b;
}