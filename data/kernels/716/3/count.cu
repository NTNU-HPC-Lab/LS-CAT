#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count(int *A, int *B, int n) {

int b_id 		= blockIdx.x,
b_num 	= gridDim.x,
b_size,
b_offset,
t_id 	= threadIdx.x,
t_num 	= blockDim.x,
t_size,
t_offset,
offset;

// initialize a shared memory array to store the count for each block.
__shared__ int count[MAX_VALUE];

// set intial values to zeros. Each thread sets its own share to zero.
t_size = (t_num > MAX_VALUE ? 1 : MAX_VALUE / t_num);
offset = t_id * t_size;
for (int i = offset; i < offset + t_size && i < MAX_VALUE; ++i)
count[i] = 0;

// wait until all threads have completed the initialization process.
__syncthreads();

// accumulate the counts of each value. Each thread counts a certain portain
// of the unsorted array.
b_size = (b_num > n ? 1 : n / b_num);
b_offset = b_id * b_size;

t_size = (t_num > b_size ? 1 : b_size / t_num);

offset = b_offset + t_id * t_size;
for (int i = offset; i < offset + t_size && i < b_offset + b_size && i < n; ++i)
atomicAdd(&count[A[i]], 1);

// wait until all threads have completed the couting phase.
__syncthreads();

// copy the block count into global memory. Each thread copies its portioin to
// the global memory.
t_size = (t_num > MAX_VALUE ? 1 : MAX_VALUE / t_num);
t_offset = t_id * t_size;
offset = b_id * MAX_VALUE + t_offset;

if (offset + t_size <= (b_id + 1) * MAX_VALUE)
memcpy(&B[offset], &count[t_offset], sizeof(int) * t_size);

}