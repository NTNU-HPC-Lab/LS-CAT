#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
long i = start;
long j = middle;
for (long k = start; k < end; k++) {
if (i < middle && (j >= end || source[i] < source[j])) {
dest[k] = source[i];
i++;
} else {
dest[k] = source[j];
j++;
}
}
}
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
int x;
return threadIdx.x +
threadIdx.y * (x  = threads->x) +
threadIdx.z * (x *= threads->y) +
blockIdx.x  * (x *= threads->z) +
blockIdx.y  * (x *= blocks->z) +
blockIdx.z  * (x *= blocks->y);
}
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
unsigned int idx = getIdx(threads, blocks);
long start = width*idx*slices,
middle,
end;

for (long slice = 0; slice < slices; slice++) {
if (start >= size)
break;

middle = min(start + (width >> 1), size);
end = min(start + width, size);
gpu_bottomUpMerge(source, dest, start, middle, end);
start += width;
}
}