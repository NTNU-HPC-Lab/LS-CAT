#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitonic_sort_step(int *dev_values, int j, int k)
{
unsigned int i, ixj; /* Sorting partners: i and ixj */
i = threadIdx.x + blockDim.x * blockIdx.x;
ixj = i^j;

/* The threads with the lowest ids sort the array. */
if ((ixj)>i) {
if ((i&k)==0) {
/* Sort ascending */
if (dev_values[i]>dev_values[ixj]) {
// swap
int temp = dev_values[i];
dev_values[i] = dev_values[ixj];
dev_values[ixj] = temp;
}
}
if ((i&k)!=0) {
/* Sort descending */
if (dev_values[i]<dev_values[ixj]) {
// swap
int temp = dev_values[i];
dev_values[i] = dev_values[ixj];
dev_values[ixj] = temp;
}
}
}
}