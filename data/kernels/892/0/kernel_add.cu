#include "hip/hip_runtime.h"
#include "includes.h"



struct MPIGlobalState {
// The CUDA device to run on, or -1 for CPU-only.
int device = -1;

// A CUDA stream (if device >= 0) initialized on the device
hipStream_t stream;

// Whether the global state (and MPI) has been initialized.
bool initialized = false;
};

// MPI relies on global state for most of its internal operations, so we cannot
// design a library that avoids global state. Instead, we centralize it in this
// single global struct.
static MPIGlobalState global_state;

// Initialize the library, including MPI and if necessary the CUDA device.
// If device == -1, no GPU is used; otherwise, the device specifies which CUDA
// device should be used. All data passed to other functions must be on that device.
//
// An exception is thrown if MPI or CUDA cannot be initialized.
__global__ void kernel_add(const float* x, const float* y, const int N, float* out) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
out[i] = x[i] + y[i];
}
}