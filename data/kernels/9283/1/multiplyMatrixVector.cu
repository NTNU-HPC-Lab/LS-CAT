#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This file is an attempt at producing what the generated target code
* should look like for the multiplyMatrixMatrix routine.
*/

/* Prototype matrix representation. */
struct dag_array_t{
size_t rows;
size_t cols;
int* matrix;
};

/*
DAG Primitive. Here, we leverage the NVIDIA developer examples
to obtain a high-bandwith operation. They make use of shared memory
to avoid strided global memory accesses, and instead perform the
strided access in the shared block, which is roughly a ~3x improvement.

TILE_DIM = 32
BLOCK_ROWS = 8

https://devblogs.nvidia.com/efficient-matrix-transpose-cuda-cc/
*/
const int tp_TILE_DIM = 32;
const int tp_BLOCK_ROWS = 8;




// We use single-dimensional lists.
__global__ void multiplyMatrixVector(int* result, int* matrix, int* vector, int cols)
{
__shared__ int reduce_array[256]; // Within a block

int vector_slice_offset = blockIdx.x * cols + threadIdx.x;
int matrix_slice_offset = blockIdx.y * cols + threadIdx.x;
reduce_array[threadIdx.x] = matrix[matrix_slice_offset] * vector[vector_slice_offset];

__syncthreads();

// Sequential reduce.
if (threadIdx.x == 0){
int accumulator = 0;
for (int i = 0; i < blockDim.x; i++)
{
accumulator += reduce_array[i];
}
result[blockIdx.x * cols + blockIdx.y] = accumulator;
}
}