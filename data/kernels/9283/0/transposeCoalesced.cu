#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This file is an attempt at producing what the generated target code
* should look like for the multiplyMatrixMatrix routine.
*/

/* Prototype matrix representation. */
struct dag_array_t{
size_t rows;
size_t cols;
int* matrix;
};

/*
DAG Primitive. Here, we leverage the NVIDIA developer examples
to obtain a high-bandwith operation. They make use of shared memory
to avoid strided global memory accesses, and instead perform the
strided access in the shared block, which is roughly a ~3x improvement.

TILE_DIM = 32
BLOCK_ROWS = 8

https://devblogs.nvidia.com/efficient-matrix-transpose-cuda-cc/
*/
const int tp_TILE_DIM = 32;
const int tp_BLOCK_ROWS = 8;




// We use single-dimensional lists.
__global__ void transposeCoalesced(int *result, const int *in)
{
const int TILE_DIM = tp_TILE_DIM;
const int BLOCK_ROWS = tp_BLOCK_ROWS;

__shared__ int tile[TILE_DIM][TILE_DIM];

int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
tile[threadIdx.y+j][threadIdx.x] = in[(y+j)*width + x];

__syncthreads();

x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
y = blockIdx.x * TILE_DIM + threadIdx.y;

for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
result[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}