#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
const auto thread_index =
threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
if (thread_index >= group_count) {
return;
}
for (size_t i = 0; i < col_count; groups += col_widths[i++] * group_count) {
switch (col_widths[i]) {
case 4:
*reinterpret_cast<uint32_t*>(groups) =
*reinterpret_cast<const uint32_t*>(init_vals + i);
break;
case 8:
reinterpret_cast<size_t*>(groups)[thread_index] = init_vals[i];
break;
default:;
}
}
}