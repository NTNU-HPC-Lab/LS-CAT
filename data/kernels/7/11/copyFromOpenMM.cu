#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyFromOpenMM( float *target, float *source, int N ) {
const int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
if( elementNum > N ) {
return;
}

const int atom = elementNum / 3;
target[elementNum] = source[4 * atom + elementNum % 3];
}