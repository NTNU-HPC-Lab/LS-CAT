#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void orthogonalize23( float *Qi_gdof, int *blocksizes, int numblocks, int largestblock ) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
for( int j = 4; j < 6; j++ ) {
for( int k = 3; k < j; k++ ) { // <-- vectors we're orthognalizing against
float dot_prod = 0.0;
for( int l = 0; l < blocksizes[i]; l++ ) {
dot_prod += Qi_gdof[i * 6 * largestblock + l * 6 + k] * Qi_gdof[i * 6 * largestblock + l * 6 + j];
}
//dot_prod += Qi_gdof[i][l][k] * Qi_gdof[i][l][j];
for( int l = 0; l < blocksizes[i]; l++ ) {
Qi_gdof[i * 6 * largestblock + l * 6 + j] -= Qi_gdof[i * 6 * largestblock + l * 6 + k] * dot_prod;
}
//Qi_gdof[i][l][j] -= Qi_gdof[i][l][k] * dot_prod;
}

float rotnorm = 0.0;
for( int l = 0; l < blocksizes[i]; l++ ) {
rotnorm += Qi_gdof[i * 6 * largestblock + l * 6 + j] * Qi_gdof[i * 6 * largestblock + l * 6 + j];
}
//rotnorm += Qi_gdof[i][l][j] * Qi_gdof[i][l][j];

rotnorm = 1.0 / sqrt( rotnorm );

for( int l = 0; l < blocksizes[i]; l++ ) {
Qi_gdof[i * 6 * largestblock + l * 6 + j] *= rotnorm;
}
//Qi_gdof[i][l][j] *= rotnorm;
}
}