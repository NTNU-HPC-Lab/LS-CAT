#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeProjection( float *eT, float *e, float *eigenvec, int *indices, int M, int N ) {
int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
if( elementNum >= M * N ) {
return;
}
int m = elementNum / N;
int n = elementNum % N;
e[n * M + m] = eigenvec[n * M + indices[m]];
eT[m * N + n] = e[n * M + m];
}