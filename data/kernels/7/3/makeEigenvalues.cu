#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeEigenvalues( float *eigenvalues, float *blockHessian, int *blocknums, int *blocksizes, int *hessiannums, int N, int numblocks ) {
// elementnum is the degree of freedom (0 to 3n-1)
int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
if( elementNum >= N ) {
return;
}

// b is the block number in which DOF elementnum resides
// blocknums contains atom numbers, so we must divide by 3
// We find the first index with an atom number larger than
// ours, and take one less (or numblocks-1 if we are at the end)
int b = 0;
while( b < numblocks ) {
if( blocknums[b] > elementNum / 3 ) {
break;
}
b++;
}
b--;

// 3*blocknums[b] is the starting degree of freedom for our block
// We must compute an offset from that, call it x.
int x = elementNum - 3 * blocknums[b];

// We initialize our spot to hessiannums[b], which is the starting
// Hessian location for our block.
// We then want to take the diagonal entry from that offset
// So element (x,x)
int spot = hessiannums[b] + x * ( 3 * blocksizes[b] ) + x;

eigenvalues[elementNum] = blockHessian[spot];
}