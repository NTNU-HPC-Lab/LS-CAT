#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMulKernel( float *C, float *A, float *B, int Aheight, int Awidth, int Bwidth ) {
float result = 0;
int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
if( elementNum > Aheight * Bwidth ) {
return;
}
int row = elementNum / Bwidth;
int col = elementNum % Bwidth;
for( int e = 0; e < Awidth; e++ ) {
result += A[row * Awidth + e] * B[e * Bwidth + col];
}
C[row * Bwidth + col] = result;
}