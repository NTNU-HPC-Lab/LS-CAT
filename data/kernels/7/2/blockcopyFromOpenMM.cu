#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blockcopyFromOpenMM( float *target, float *source, int *blocks, int numblocks, int setnum, int N ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
int dof = 3 * blocks[blockNum] + setnum;
int atom = dof / 3;

if( atom >= N || ( blockNum != numblocks && atom >= blocks[blockNum + 1] ) ) {
return;    // Out of bounds
}

target[dof] = *( source + ( dof + atom + 1 ) * sizeof( float ) ); // Save the old
}