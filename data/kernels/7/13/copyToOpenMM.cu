#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyToOpenMM( float *target, float *source, int N ) {
int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
int atom = elementNum / 3;
if( elementNum > N ) {
return;
}
//else target[elementNum] = source[elementNum];
else {
target[4 * atom + elementNum % 3] = source[elementNum];
}
}