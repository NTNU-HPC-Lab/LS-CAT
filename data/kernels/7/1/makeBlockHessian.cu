#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeBlockHessian( float *h, float *forces1, float *forces2, float *mass, float blockDelta, int *blocks, int *blocksizes, int numblocks, int *hessiannums, int *hessiansizes, int setnum, int N ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
int dof = 3 * blocks[blockNum] + setnum;
int atom = dof / 3;
if( atom >= N || ( blockNum != numblocks - 1 && atom >= blocks[blockNum + 1] ) ) {
return;    // Out of bounds
}

int start_dof = 3 * blocks[blockNum];
int end_dof;
if( blockNum == numblocks - 1 ) {
end_dof = 3 * N;
} else {
end_dof = 3 * blocks[blockNum + 1];
}

/* I also would like to parallelize this at some point as well */
for( int k = start_dof; k < end_dof; k++ ) {
float blockScale = 1.0 / ( blockDelta * sqrt( mass[atom] * mass[k / 3] ) );
//h[startspot+i] = (forces1[k] - forces2[k]) * blockScale;
h[hessiannums[blockNum] + ( k - start_dof ) * ( 3 * blocksizes[blockNum] ) + ( dof - start_dof )] = ( forces1[k] - forces2[k] ) * blockScale;
}
}