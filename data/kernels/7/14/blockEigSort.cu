#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blockEigSort( float *eigenvalues, float *eigenvectors, int *blocknums, int *blocksizes, int N ) {
int blockNumber = blockIdx.x * blockDim.x + threadIdx.x;
int startspot = blocknums[blockNumber];
int endspot = startspot + blocksizes[blockNumber] - 1;

// Bubble sort for now, thinking blocks are relatively small
// We may fix it later
for( int i = startspot; i < endspot; i++ ) {
for( int j = startspot; j < i; j++ ) {
if( eigenvalues[j] > eigenvalues[j + 1] ) {
float tmp = eigenvalues[j];
eigenvalues[j] = eigenvalues[j + 1];
eigenvalues[j + 1] = tmp;

// Swapping addresses
for( int i = 0; i < N; i++ ) {
tmp = eigenvectors[i * N + j];
eigenvectors[i * N + j] = eigenvectors[i * N + j + 1];
eigenvectors[i * N + j + 1] = tmp;
}
/*float* tmpaddr = eigenvectors[j];
eigenvectors[j] = eigenvectors[j+1];;
eigenvectors[j+1] = tmpaddr;*/
}
}
}
}