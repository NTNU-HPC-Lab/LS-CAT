#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void symmetrize1D( float *h, int *blockPositions, int *blockSizes, int numBlocks ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
if( blockNum >= numBlocks ) {
return;
}

// blockSizes are given in terms of atoms, convert to dof
const unsigned int blockSize = 3 * blockSizes[blockNum];

float *block = &( h[blockPositions[blockNum]] );
for( unsigned int r = 0; r < blockSize - 1; r++ ) {
for( unsigned int c = r + 1; c < blockSize; c++ ) {
const float avg = 0.5f * ( block[r * blockSize + c] + block[c * blockSize + r] );
block[r * blockSize + c] = avg;
block[c * blockSize +	r] = avg;
}
}
}