#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void symmetrize2D( float *h, int natoms ) {
const int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
const int dof = 3 * natoms;
if( elementNum >= dof * dof ) {
return;
}
int r = elementNum / dof;
int c = elementNum % dof;

if( r > c ) {
return;
} else {
const float avg = 0.5 * ( h[r * dof + c] + h[c * dof + r] );
h[r * dof + c] = avg;
h[c * dof + r] = avg;
}
}