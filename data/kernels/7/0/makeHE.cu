#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeHE( float *HE, float *force1, float4 *force2, float *masses, float eps, int k, int m, int N ) {
int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
int atom = elementNum / 3;
if( elementNum >= N ) {
return;
}

int axis = elementNum % 3;
if( axis == 0 ) {
HE[elementNum * m + k] = ( force1[elementNum] - force2[atom].x ) / ( sqrt( masses[atom] ) * 1.0 * eps );
} else if( axis == 1 ) {
HE[elementNum * m + k] = ( force1[elementNum] - force2[atom].y ) / ( sqrt( masses[atom] ) * 1.0 * eps );
} else {
HE[elementNum * m + k] = ( force1[elementNum] - force2[atom].z ) / ( sqrt( masses[atom] ) * 1.0 * eps );
}
}