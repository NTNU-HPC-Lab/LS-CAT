#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void geometricDOF( float *Qi_gdof, float4 *positions, float *masses, int *blocknums, int *blocksizes, int largestsize, float *norm, float *pos_center ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
for( int j = 0; j < blocksizes[blockNum] - 3; j += 3 ) {

int atom = ( blocknums[blockNum] + j ) / 3;
float mass = masses[atom];
float factor = sqrt( mass ) / norm[atom];

Qi_gdof[blockNum * largestsize * 6 + j * 6 + 0]   = factor;
Qi_gdof[blockNum * largestsize * 6 + ( j + 1 ) * 6 + 1] = factor;
Qi_gdof[blockNum * largestsize * 6 + ( j + 2 ) * 6 + 2] = factor;

float diff0 = positions[atom].x - pos_center[atom * 3 + 0];
float diff1 = positions[atom].y - pos_center[atom * 3 + 1];
float diff2 = positions[atom].z - pos_center[atom * 3 + 2];

Qi_gdof[blockNum * largestsize * 6 + ( j + 1 ) * 6 + 3] = diff2 * factor;
Qi_gdof[blockNum * largestsize * 6 + ( j + 2 ) * 6 + 3] = -diff1 * factor;

Qi_gdof[blockNum * largestsize * 6 + ( j ) * 6 + 4] = -diff2 * factor;
Qi_gdof[blockNum * largestsize * 6 + ( j + 2 ) * 6 + 4] = -diff0 * factor;

Qi_gdof[blockNum * largestsize * 6 + ( j ) * 6 + 5] = diff1 * factor;
Qi_gdof[blockNum * largestsize * 6 + ( j + 1 ) * 6 + 5] = -diff0 * factor;
}
// Normalize first vector
float rotnorm = 0.0;
for( int j = 0; j < blocksizes[blockNum]; j++ ) {
rotnorm += Qi_gdof[blockNum * largestsize * 6 + j * 6 + 3] * Qi_gdof[blockNum * largestsize * 6 + j * 6 + 3];
}

rotnorm = 1.0 / sqrt( rotnorm );

for( int j = 0; j < blocksizes[blockNum]; j++ ) {
Qi_gdof[blockNum * largestsize * 6 + j * 6 + 3] *= rotnorm;
}
}