#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void orthogonalize( float *eigvec, float *Qi_gdof, int cdof, int *blocksizes, int *blocknums, int largestblock ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;

// orthogonalize original eigenvectors against gdof
// number of evec that survive orthogonalization
int curr_evec = 6;
int size = blocksizes[blockNum];
int startatom = blocknums[blockNum] / 3;
for( int j = 0; j < size; j++ ) { // <-- vector we're orthogonalizing
// to match ProtoMol we only include size instead of size + cdof vectors
// Note: for every vector that is skipped due to a low norm,
// we add an additional vector to replace it, so we could actually
// use all size original eigenvectors
if( curr_evec == size ) {
break;
}

// orthogonalize original eigenvectors in order from smallest magnitude
// eigenvalue to biggest
// TMC The eigenvectors are sorted now
//int col = sortedPairs.at( j ).second;

// copy original vector to Qi_gdof -- updated in place
for( int l = 0; l < size; l++ ) {
//Qi_gdof[blockNum*6*largestblock+l*6+curr_evec] = eigvec[blocknums[blockNum]+l][j];
Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] = eigvec[( blocknums[blockNum] + l ) * largestblock + j];
}

// get dot products with previous vectors
for( int k = 0; k < curr_evec; k++ ) { // <-- vector orthog against
// dot product between original vector and previously
// orthogonalized vectors
double dot_prod = 0.0;
for( int l = 0; l < size; l++ ) {
//dot_prod += Qi_gdof[blockNum*6*largestblock+l*6+k] * eigvec[blocknums[blockNum]+l][j];
dot_prod += Qi_gdof[blockNum * 6 * largestblock + l * 6 + k] * eigvec[( blocknums[blockNum] + l ) * largestblock + j];
}

// subtract from current vector -- update in place
for( int l = 0; l < size; l++ ) {
Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] = Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] - Qi_gdof[blockNum * 6 * largestblock + l * 6 + k] * dot_prod;
}
}

//normalize residual vector
double norm = 0.0;
for( int l = 0; l < size; l++ ) {
norm += Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] * Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec];
}

// if norm less than 1/20th of original
// continue on to next vector
// we don't update curr_evec so this vector
// will be overwritten
if( norm < 0.05 ) {
continue;
}

// scale vector
norm = sqrt( norm );
for( int l = 0; l < size; l++ ) {
Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] = Qi_gdof[blockNum * 6 * largestblock + l * 6 + curr_evec] / norm;
}

curr_evec++;
}

// 4. Copy eigenpairs to big array
//    This is necessary because we have to sort them, and determine
//    the cutoff eigenvalue for everybody.
// we assume curr_evec <= size
for( int j = 0; j < curr_evec; j++ ) {
//eval[startatom + j] = di[col]; No longer necessary

// orthogonalized eigenvectors already sorted by eigenvalue
for( int k = 0; k < size; k++ ) {
//eigvec[startatom + k][startatom + j] = Qi_gdof[blockNum*6*largestblock+k*6+j];
eigvec[( startatom + k )*largestblock + ( startatom + j )] = Qi_gdof[blockNum * 6 * largestblock + k * 6 + j];
}
}
}