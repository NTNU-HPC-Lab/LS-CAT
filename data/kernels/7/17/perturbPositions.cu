#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void perturbPositions( float *blockpos, float4 *initpos, float delta, int *blocks, int numblocks, int setnum, int N ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;

if( blockNum >= numblocks ) {
return;
}

int dof = 3 * blocks[blockNum] + setnum;
int atom = dof / 3;

if( atom >= N || ( blockNum != ( numblocks - 1 ) && atom >= blocks[blockNum + 1] ) ) {
return;    // Out of bounds
}

int axis = dof % 3;
if( axis == 0 ) {
blockpos[dof] = initpos[atom].x;
initpos[atom].x += delta;
} else if( axis == 1 ) {
blockpos[dof] = initpos[atom].y;
initpos[atom].y += delta;
} else {
blockpos[dof] = initpos[atom].z;
initpos[atom].z += delta;
}
}