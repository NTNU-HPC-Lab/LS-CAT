#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeNormsAndCenter( float *norms, float *center, float *masses, float4 *positions, int *blocknums, int *blocksizes ) {
int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
float totalmass = 0.0;
for( int j = blocknums[blockNum]; j <= blocknums[blockNum] + blocksizes[blockNum] - 1; j += 3 ) {
float mass = masses[ j / 3 ];
center[blockNum * 3 + 0] = positions[j / 3].x * mass;
center[blockNum * 3 + 1] = positions[j / 3].y * mass;
center[blockNum * 3 + 2] = positions[j / 3].z * mass;
totalmass += mass;
}

norms[blockNum] = sqrt( totalmass );
center[blockNum * 3 + 0] /= totalmass;
center[blockNum * 3 + 1] /= totalmass;
center[blockNum * 3 + 2] /= totalmass;
}