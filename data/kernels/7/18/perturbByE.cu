#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void perturbByE( float *tmppos, float4 *mypos, float eps, float *E, float *masses, int k, int m, int N ) {
int dof = blockIdx.x * blockDim.x + threadIdx.x;
if( dof >= N ) {
return;
}
int atom = dof / 3;

int axis = dof % 3;
if( axis == 0 ) {
tmppos[dof] = mypos[atom].x;
mypos[atom].x += eps * E[dof * m + k] / sqrt( masses[atom] );
} else if( axis == 1 ) {
tmppos[dof] = mypos[atom].y;
mypos[atom].y += eps * E[dof * m + k] / sqrt( masses[atom] );
} else {
tmppos[dof] = mypos[atom].z;
mypos[atom].z += eps * E[dof * m + k] / sqrt( masses[atom] );
}
}