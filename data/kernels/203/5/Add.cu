#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float *A, int size)
{
const unsigned int numThreads = blockDim.x * gridDim.x;
const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

for (unsigned int i = idx;i < size; i += numThreads)
A[i] = A[i]+ A[i];
}