#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixTranspose(float *a,float *b,int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;
int odx= iy*nx + ix;

if((ix<nx)&&(iy<ny)){
b[odx]=a[idx];
}

}