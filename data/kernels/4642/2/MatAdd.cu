#include "hip/hip_runtime.h"
#include "includes.h"








__global__ void MatAdd(float *A, float *B, float *C, int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;

if((ix<nx)&&(iy<ny)){
C[idx]=A[idx]+B[idx];
}

}