#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 40

//-----------------------------------------------



//--------------------------------------------------

// Compute C = A * B

//-------------------------------------------------

__global__ void MatrixMult(int m, int n, int k, double *a, double *b, double *c)
{

int row = threadIdx.y + blockIdx.y*blockDim.y;
int col = threadIdx.x + blockIdx.x*blockDim.x;

if((row < m) && (col < k))
{
double temp = 0.0;
for (int i = 0; i < n; ++i)
{
temp += a[row*n+i]*b[col+i*k];
}
c[row*k+col] = temp;
}

}