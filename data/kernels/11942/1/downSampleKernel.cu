#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void downSampleKernel(unsigned char * d_in, unsigned char * d_out, size_t skip) {
size_t i = threadIdx.x;
// Assuming 3 channels BGR and averaging
int px = d_in[i * skip * 3] + d_in[i * skip * 3 + 1] + d_in[i * skip * 3 + 2];
d_out[i] = px / 3;
}