#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This code is released into the public domain.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
* OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
* ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
* OTHER DEALINGS IN THE SOFTWARE.
*/






///////////////////////////////////////////////////////////////////////////////////////////
// Definitions and helper utilities

// Block width for CUDA kernels
#define BW 128

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
__global__ void FillOnes(float *vec, int size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= size)
return;

vec[idx] = 1.0f;
}