#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This code is released into the public domain.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
* OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
* ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
* OTHER DEALINGS IN THE SOFTWARE.
*/






///////////////////////////////////////////////////////////////////////////////////////////
// Definitions and helper utilities

// Block width for CUDA kernels
#define BW 128

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx >= batch_size)
return;

const int label_value = static_cast<int>(label[idx]);

// For each item in the batch, decrease the result of the label's value by 1
diff[idx * num_labels + label_value] -= 1.0f;
}