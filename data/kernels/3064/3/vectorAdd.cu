#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(int* a, int* b, int* c, int n){
int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < n){
c[tid] = a[tid] + b[tid];
}

}