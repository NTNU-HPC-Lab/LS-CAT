#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *a, int *b, int *c, int n, int tile_size){
__shared__ int A[SHMEM_SIZE];
__shared__ int B[SHMEM_SIZE];

int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x;
int by = blockIdx.y;

int row = by * tile_size + ty;
int col = bx * tile_size + tx;

int temp_sum = 0;

for (int i = 0; i < (n / tile_size); i++){
A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col];

__syncthreads();

for(int j = 0; j < tile_size; j++){
temp_sum += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
}

__syncthreads();
}

c[(row * n) + col] = temp_sum;
}