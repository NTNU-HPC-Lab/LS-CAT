#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *a, int *b, int *c, int n){
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

int temp_sum = 0;

if((row < n) && (col < n)){
for (int k = 0; k < n; k++){
temp_sum += a[row * n + k] * b[k * n + col];
}

c[row * n + col] = temp_sum;
}

}