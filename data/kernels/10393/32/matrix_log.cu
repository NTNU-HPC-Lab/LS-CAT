#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/


/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: rows of output matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: subset of number of non-zeroes of input matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is dense and the output matrix is dense.
*
* @params in dense input pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param inClen number of columns of input matrix
* @param retRlen number of rows of output matrix
* @param retClen number of columns of output matrix
*/
extern "C"


/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

extern "C"
__global__ void matrix_log(double *A, double *C, unsigned int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size){
C[index] = log(A[index]);
}
}