#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePressure_CUDA(float* pressure, float* density, const int num, const float rho0, const float stiff)
{
const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
if (i >= num) return;
pressure[i] = stiff * (powf((density[i] / rho0), 7) - 1.0f);
//clamp
if (pressure[i] < 0.0f) pressure[i] = 0.0f;
return;
}