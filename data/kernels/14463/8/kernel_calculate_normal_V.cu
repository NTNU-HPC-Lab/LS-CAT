#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Quantum Lattice Boltzmann
*  (c) 2015 Fabian Thüring, ETH Zurich
*
*  This file contains all the CUDA kernels and function that make use of the
*  CUDA runtime API
*/

// Local includes

// ==== CONSTANTS ====

__constant__ unsigned int d_L;
__constant__ float d_dx;
__constant__ float d_dt;
__constant__ float d_mass;
__constant__ float d_g;
__constant__ unsigned int d_t;

__constant__ float d_scaling;
__constant__ int d_current_scene;

// ==== INITIALIZATION ====

__global__ void kernel_calculate_normal_V(float3* vbo_ptr, float* d_ptr)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if(i < d_L && j < d_L)
{
int ik = (i + 1) % d_L;
int jk = (d_L - 1 + j) % d_L;

// x
float x2 =  d_scaling * fabsf( d_ptr[i*d_L +j] );

// a
float a1 =  d_dx;
float a2 =  d_scaling * fabsf( d_ptr[ik*d_L +j] ) - x2;

// b
float b2 =  d_scaling * fabsf( d_ptr[i*d_L +jk] ) - x2;
float b3 = -d_dx;

// n = a x b
float3 n;
n.x =  a2*b3;
n.y = -a1*b3;
n.z =  a1*b2;

// normalize
float norm = sqrtf(n.x*n.x + n.y*n.y + n.z*n.z);

vbo_ptr[d_L*i + j].x = n.x/norm;
vbo_ptr[d_L*i + j].y = n.y/norm;
vbo_ptr[d_L*i + j].z = n.z/norm;
}
}