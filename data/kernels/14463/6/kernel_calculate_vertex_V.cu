#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Quantum Lattice Boltzmann
*  (c) 2015 Fabian Thüring, ETH Zurich
*
*  This file contains all the CUDA kernels and function that make use of the
*  CUDA runtime API
*/

// Local includes

// ==== CONSTANTS ====

__constant__ unsigned int d_L;
__constant__ float d_dx;
__constant__ float d_dt;
__constant__ float d_mass;
__constant__ float d_g;
__constant__ unsigned int d_t;

__constant__ float d_scaling;
__constant__ int d_current_scene;

// ==== INITIALIZATION ====

__global__ void kernel_calculate_vertex_V(float3* vbo_ptr, float* d_ptr)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;

if(i < d_L && j < d_L)
vbo_ptr[d_L*i + j].y = d_scaling * fabsf( d_ptr[i*d_L +j] ) - 0.005f*d_L;
}