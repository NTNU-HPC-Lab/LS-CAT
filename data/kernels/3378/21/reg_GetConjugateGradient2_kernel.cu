#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reg_GetConjugateGradient2_kernel(	float4 *nodeNMIGradientArray_d, float4 *conjugateG_d, float4 *conjugateH_d)
{
const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
if(tid < c_NodeNumber){
// G = - grad
float4 gradGValue = nodeNMIGradientArray_d[tid];
gradGValue = make_float4(-gradGValue.x, -gradGValue.y, -gradGValue.z, 0.0f);
conjugateG_d[tid]=gradGValue;

// H = G + gam * H
float4 gradHValue = conjugateH_d[tid];
gradHValue=make_float4(
gradGValue.x + c_ScalingFactor * gradHValue.x,
gradGValue.y + c_ScalingFactor * gradHValue.y,
gradGValue.z + c_ScalingFactor * gradHValue.z,
0.0f);
conjugateH_d[tid]=gradHValue;
nodeNMIGradientArray_d[tid]=make_float4(-gradHValue.x, -gradHValue.y, -gradHValue.z, 0.0f);
}
}