#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reg_addArrays_kernel_float(float *array1_d, float *array2_d)
{
const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
if(tid < c_VoxelNumber){
array1_d[tid] += array2_d[tid];
}
}