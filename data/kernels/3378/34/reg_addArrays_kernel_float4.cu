#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reg_addArrays_kernel_float4(float4 *array1_d, float4 *array2_d)
{
const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
if(tid < c_VoxelNumber){
float4 a = array1_d[tid];
float4 b = array1_d[tid];
array1_d[tid] = make_float4(a.x+b.x,a.y+b.y,a.z+b.z,a.w+b.w);
}
}