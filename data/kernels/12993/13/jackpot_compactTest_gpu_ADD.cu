#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jackpot_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
__shared__ uint32_t buf;
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

if (id > len) return;

if (threadIdx.x == 0)
{
buf = partial_sums[blockIdx.x];
}

__syncthreads();
data[id] += buf;
}