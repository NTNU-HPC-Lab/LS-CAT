#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void magnitudeImage(float *d_gradientX, float *d_gradientY, float *d_gradientMag, int width, int height) {
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < width && iy < height) {
int idx = iy * width + ix;
d_gradientMag[idx] = sqrtf(powf(d_gradientX[idx], 2.0f) + powf(d_gradientY[idx], 2.0f));
}
}