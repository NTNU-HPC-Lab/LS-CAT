#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void initializeSobel(float *d_sobelKernelX, float *d_sobelKernelY) {
int ix = threadIdx.x;
int iy = threadIdx.y;
int weight = SOBEL_KERNEL_SIZE / 2;

if (ix < SOBEL_KERNEL_SIZE && iy < SOBEL_KERNEL_SIZE) {
int index = iy * SOBEL_KERNEL_SIZE + ix;
float sx = ix - SOBEL_KERNEL_SIZE / 2;
float sy = iy - SOBEL_KERNEL_SIZE / 2;
float norm = sx * sx + sy *sy;

if (norm == 0.0f) {
d_sobelKernelX[index] = 0.0f;
d_sobelKernelY[index] = 0.0f;
}
else {
d_sobelKernelX[index] = sx * weight / norm;
d_sobelKernelY[index] = sy * weight / norm;
}
}
}