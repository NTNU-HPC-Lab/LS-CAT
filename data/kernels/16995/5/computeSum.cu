#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void computeSum(float *d_filteredImage, float *d_imageSumGrid, unsigned int n)
{
__shared__ float smem[SMEM_SIZE];
unsigned int tid = threadIdx.x;
unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
float localSum = 0;

if (idx + 3 * blockDim.x < n)
{
float a1 = d_filteredImage[idx];
float a2 = d_filteredImage[idx + blockDim.x];
float a3 = d_filteredImage[idx + 2 * blockDim.x];
float a4 = d_filteredImage[idx + 3 * blockDim.x];
localSum = a1 + a2 + a3 + a4;
}

smem[tid] = localSum;
__syncthreads();

if (blockDim.x >= 1024 && tid < 512)
smem[tid] += smem[tid + 512];
__syncthreads();
if (blockDim.x >= 512 && tid < 256)
smem[tid] += smem[tid + 256];
__syncthreads();
if (blockDim.x >= 256 && tid < 128)
smem[tid] += smem[tid + 128];
__syncthreads();
if (blockDim.x >= 128 && tid < 64)
smem[tid] += smem[tid + 64];
__syncthreads();

if (tid < 32)
{
volatile float *vsmem = smem;
vsmem[tid] += vsmem[tid + 32];
vsmem[tid] += vsmem[tid + 16];
vsmem[tid] += vsmem[tid + 8];
vsmem[tid] += vsmem[tid + 4];
vsmem[tid] += vsmem[tid + 2];
vsmem[tid] += vsmem[tid + 1];
}

if (tid == 0) d_imageSumGrid[blockIdx.x] = smem[0];
}