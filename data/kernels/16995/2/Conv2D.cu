#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void Conv2D(float *d_image, float *kernel, float *d_result, int width, int height, int kernelSize) {
const int sharedMemWidth = TILE_WIDTH + MAX(SOBEL_KERNEL_SIZE, GAUSSIAN_KERNEL_SIZE) - 1;
__shared__ float sharedMem[sharedMemWidth][sharedMemWidth];

int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
int destY = dest / sharedMemWidth;
int destX = dest % sharedMemWidth;
int srcY = blockIdx.y * TILE_WIDTH + destY - (kernelSize / 2);
int srcX = blockIdx.x * TILE_WIDTH + destX - (kernelSize / 2);
int src = (srcY * width + srcX);
if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
sharedMem[destY][destX] = d_image[src];
else
sharedMem[destY][destX] = 0;

dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
destY = dest / sharedMemWidth;
destX = dest % sharedMemWidth;
srcY = blockIdx.y * TILE_WIDTH + destY - (kernelSize / 2);
srcX = blockIdx.x * TILE_WIDTH + destX - (kernelSize / 2);
src = (srcY * width + srcX);
if (destY < sharedMemWidth) {
if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
sharedMem[destY][destX] = d_image[src];
else
sharedMem[destY][destX] = 0;
}
__syncthreads();

float accum = 0;
for (int j = 0; j < kernelSize; j++)
for (int i = 0; i < kernelSize; i++)
accum += sharedMem[threadIdx.y + j][threadIdx.x + i] * kernel[j * kernelSize + i];
int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
if (x < width && y < height)
d_result[y * width + x] = (fminf(fmaxf((accum), 0.0), 1.0));
__syncthreads();
}