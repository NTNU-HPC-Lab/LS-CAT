#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void highHysterisis(int width, int height, float* d_nonMax, float highThreshold, float *d_highThreshHyst) {
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < width && iy < height) {
int tid = iy * width + ix;

d_highThreshHyst[tid] = 0.0f;
if(d_nonMax[tid] > highThreshold)
d_highThreshHyst[tid] = 1.0f;
}
}