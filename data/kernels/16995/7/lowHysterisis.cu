#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void lowHysterisis(int width, int height, float *d_nonMax, float* d_highThreshHyst, float lowThreshold, float *d_lowThreshHyst) {
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;

if ((ix > 0) && (ix < (width - 1)) && (iy > 0) && iy < (height - 1)) {
int tid = iy * width + ix;

d_lowThreshHyst[tid] = d_highThreshHyst[tid];

if (d_highThreshHyst[tid] == 1) {
// Determine neighbour indices
int eastN = tid + 1;
int westN = tid - 1;
int northN = tid - width;
int southN = tid + width;

int southEastN = southN + 1;
int northEastN = northN	+ 1;
int southWestN = southN - 1;
int northWestN = northN	- 1;

if (d_nonMax[eastN] > lowThreshold)
d_lowThreshHyst[eastN] = 1.0f;

if (d_nonMax[westN] > lowThreshold)
d_lowThreshHyst[westN] = 1.0f;

if (d_nonMax[northN] > lowThreshold)
d_lowThreshHyst[northN] = 1.0f;

if (d_nonMax[southN] > lowThreshold)
d_lowThreshHyst[southN] = 1.0f;

if (d_nonMax[southEastN] > lowThreshold)
d_lowThreshHyst[southEastN] = 1.0f;

if (d_nonMax[northEastN] > lowThreshold)
d_lowThreshHyst[northEastN] = 1.0f;

if (d_nonMax[southWestN] > lowThreshold)
d_lowThreshHyst[southWestN] = 1.0f;

if (d_nonMax[northWestN] > lowThreshold)
d_lowThreshHyst[northWestN] = 1.0f;
}
}
}