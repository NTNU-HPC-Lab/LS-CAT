#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128
__global__ void nonMaxSuppressionDevice(int width, int height, float *d_gradientX, float *d_gradientY, float* d_gradientMag, float* d_nonMax) {
int ix = threadIdx.x + blockIdx.x * blockDim.x;
int iy = threadIdx.y + blockIdx.y * blockDim.y;

if (ix < width && iy < height) {
int tid = iy * width + ix;

float d_gradientMag_tid = d_gradientMag[tid];
float d_gradientMag_tid_next = d_gradientMag[tid + 1];
float d_gradientMag_tid_prev = d_gradientMag[tid - 1];
float d_gradientMag_tid_width_next = d_gradientMag[tid + width + 1];
float d_gradientMag_tid_width_prev = d_gradientMag[tid - width - 1];
float d_gradientMag_tid_width_plus = d_gradientMag[tid + width];
float d_gradientMag_tid_width_minus = d_gradientMag[tid - width];
float d_gradientMag_tid_width_minus_next = d_gradientMag[tid - width + 1];
float d_gradientMag_tid_width_plus_prev = d_gradientMag[tid + width - 1];

float d_gradientXT = d_gradientX[tid];
float d_gradientYT = d_gradientY[tid];

float tanYX;
float magB, magA;

if ((tid < width) || (tid >= ((height - 1) * width))) // Top and Bottom Edge
d_nonMax[tid] = 0;
else if ((tid % width == 0) || (tid % width == (width - 1))) // Left and Right Edge
d_nonMax[tid] = 0;
else {
if (d_gradientMag_tid == 0)
d_nonMax[tid] = 0;
else if (d_gradientXT >= 0) { // Direction East
if (d_gradientYT >= 0) { // Direction South-East
if (d_gradientXT >= d_gradientYT) { // East of South-East direction
tanYX = (float)(d_gradientYT / d_gradientXT);

magA = ((1 - tanYX) * d_gradientMag_tid_next) + (tanYX * d_gradientMag_tid_width_next);
magB = ((1 - tanYX) * d_gradientMag_tid_prev) + (tanYX * d_gradientMag_tid_width_plus_prev);
}
else { // South of South-East direction
tanYX = (float)(d_gradientXT / d_gradientYT);

magA = ((1 - tanYX) * d_gradientMag_tid_width_plus) + (tanYX * d_gradientMag_tid_width_next);
magB = ((1 - tanYX) * d_gradientMag_tid_width_minus) + (tanYX * d_gradientMag_tid_width_prev);
}
}
else { // Direction North-East
if (d_gradientXT >= (-1 * d_gradientYT)) { // East of North-East direction
tanYX = (float)((-1 * d_gradientYT) / d_gradientXT);

magA = ((1 - tanYX) * d_gradientMag_tid_next) + (tanYX * d_gradientMag_tid_width_minus_next);
magB = ((1 - tanYX) * d_gradientMag_tid_prev) + (tanYX * d_gradientMag_tid_width_plus_prev);
}
else { // North of North-East direction
tanYX = (float)(d_gradientXT / (-1 * d_gradientYT));

magA = ((1 - tanYX) * d_gradientMag_tid_width_plus) + (tanYX * d_gradientMag_tid_width_plus_prev);
magB = ((1 - tanYX) * d_gradientMag_tid_width_minus) + (tanYX * d_gradientMag_tid_width_minus_next);
}
}
}
else { // Direction West
if (d_gradientYT >= 0) { // Direction South-West
if (d_gradientYT >= (-1 * d_gradientXT)) { // South of South-West direction
tanYX = (float)((-1 * d_gradientXT) / d_gradientYT);
magA = ((1 - tanYX) * d_gradientMag_tid_width_plus) + (tanYX * d_gradientMag_tid_width_plus_prev);
magB = ((1 - tanYX) * d_gradientMag_tid_width_minus) + (tanYX * d_gradientMag_tid_width_minus_next);
}
else { // West of South-West direction
tanYX = (float)(d_gradientYT / (-1 * d_gradientXT));
magA = ((1 - tanYX) * d_gradientMag_tid_prev) + (tanYX * d_gradientMag_tid_width_plus_prev);
magB = ((1 - tanYX) * d_gradientMag_tid_next) + (tanYX * d_gradientMag_tid_width_minus_next);
}
}
else { // Direction North-West
if (d_gradientYT >= d_gradientXT) { // West of North-West direction
tanYX = (float)(d_gradientYT / d_gradientXT);
magA = ((1 - tanYX) * d_gradientMag_tid_prev) + (tanYX * d_gradientMag_tid_width_prev);
magB = ((1 - tanYX) * d_gradientMag_tid_next) + (tanYX * d_gradientMag_tid_width_next);
}
else {// North of North-West direction
tanYX = (float)(d_gradientXT / d_gradientYT);
magA = ((1 - tanYX) * d_gradientMag_tid_width_plus) + (tanYX * d_gradientMag_tid_width_next);
magB = ((1 - tanYX) * d_gradientMag_tid_width_minus) + (tanYX * d_gradientMag_tid_width_prev);
}
}
}

if ((d_gradientMag_tid < magA) || (d_gradientMag_tid < magB))
d_nonMax[tid] = 0;
else
d_nonMax[tid] = d_gradientMag_tid;
}
}
}