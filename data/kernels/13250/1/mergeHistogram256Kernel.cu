#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeHistogram256Kernel( uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount )
{
uint sum = 0;

for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
{
sum += d_PartialHistograms[blockIdx.x + i * HISTO256_BINS];
}

__shared__ uint data[MERGE_THREADBLOCK_SIZE];
data[threadIdx.x] = sum;

for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
{
__syncthreads();

if (threadIdx.x < stride)
{
data[threadIdx.x] += data[threadIdx.x + stride];
}
}

if (threadIdx.x == 0)
{
d_Histogram[blockIdx.x] = data[0];
}
}