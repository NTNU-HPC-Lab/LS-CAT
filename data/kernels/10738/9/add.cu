#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int* b, int* c) {
int id = blockIdx.x * blockDim.x + threadIdx.x;
c[id] = a[id] + b[id];
}