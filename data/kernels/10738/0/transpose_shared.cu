#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// parameter describing the size of matrix A
const int rows = 4096;
const int cols = 4096;

const int BLOCK_SIZE = 16;

// transpose shared kernel

// transpose kernel

__global__ void transpose_shared(float* a, float*b) {
__shared__ float result[BLOCK_SIZE][BLOCK_SIZE+1];
// adding one to avoid bank conflict

int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int width = gridDim.x * blockDim.x;
int height = gridDim.y * blockDim.y;

// perform transpose
if (x < height && y < width) {
result[threadIdx.x][threadIdx.y] = a[y*height + x];
}
__syncthreads();
b[x*height + y] = result[threadIdx.x][threadIdx.y];
}