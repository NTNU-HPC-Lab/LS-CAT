#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c,int columns,int rows)
{
// get the global id for the thread
int x = (blockIdx.x * blockDim.x + threadIdx.x);
int y = (blockIdx.y * blockDim.y + threadIdx.y);

// calculate the index of the input data
int index = y * columns + x;

c[index] = a[index] + b[index];
}