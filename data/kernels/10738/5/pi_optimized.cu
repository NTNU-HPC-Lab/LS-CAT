#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pi_optimized(float* x, float* y, int* global_count) {
__shared__ int counts[nthreads];

//int globalId = blockIdx.x * blockDim.x + nitemsperthread * threadIdx.x;
int globalId = blockIdx.x * blockDim.x + threadIdx.x;

int thread_count=0;
for (int i=0; i<nitemsperthread; i++) {
int idx = globalId+(i*nthreads*nblocks);
if (idx < nsamples) {
if (x[idx]*x[idx] + y[idx]*y[idx] < 1.0) {
thread_count++;
}
}
}

counts[threadIdx.x] = thread_count;
__syncthreads();

if (threadIdx.x == 0) {
int block_count = 0;
for (int i=0; i<nthreads; i++) {
block_count += counts[i];
}
global_count[blockIdx.x] = block_count;
}
}