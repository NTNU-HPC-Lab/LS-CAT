#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pi_random(float* x, float* y, int* global_count) {
__shared__ int counts[nthreads];

int globalId = blockIdx.x * blockDim.x + nitemsperthread * threadIdx.x;

int thread_count=0;
for (int i=0; i<nitemsperthread; i++) {
if (globalId+i < nsamples) {
if (x[globalId+i]*x[globalId+i] + y[globalId+i]*y[globalId+i] < 1.0) {
thread_count++;
}
}
}

counts[threadIdx.x] = thread_count;
__syncthreads();

if (threadIdx.x == 0) {
int block_count = 0;
for (int i=0; i<nthreads; i++) {
block_count += counts[i];
}
global_count[blockIdx.x] = block_count;
}
}