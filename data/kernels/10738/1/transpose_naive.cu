#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// parameter describing the size of matrix A
const int rows = 4096;
const int cols = 4096;

const int BLOCK_SIZE = 16;

// transpose shared kernel

// transpose kernel

__global__ void transpose_naive(float* a, float*b) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
int width = gridDim.x * blockDim.x;
int height = gridDim.y * blockDim.y;

// perform transpose
if (x < height && y < width) {
b[x*height + y] = a[y*width + x];
}
}