#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dropout_op(size_t sz, float_t* random_nums, float_t* data, float_t drop_rate, float_t scale)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
if(index < sz)
{
if(random_nums[index] <= drop_rate)
{
data[index] = 0;
}
else
{
data[index] *= scale;
}
}
}