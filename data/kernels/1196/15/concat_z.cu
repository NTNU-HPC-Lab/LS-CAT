#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void concat_z(size_t sz, float_t* src, float_t* dest, float_t* z, size_t stride)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

if(index < sz)
{
if(index>=stride)
{
dest[index]=src[index-stride];
}
else
{
dest[index]=z[index];
}
}
}