#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transformation(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;

if(index < num_values)
{
size_t dest_index = (index/ld_src)*ld_src + ((index%ld_src)%8)*ld_dest+ (index%ld_src)/8;
dest[dest_index] = src[index];
}
}