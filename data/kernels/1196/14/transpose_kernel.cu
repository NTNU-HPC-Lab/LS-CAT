#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_kernel(size_t sz, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
size_t i = index/ld_src, j= index%ld_src;
size_t dest_index = j*ld_dest + i;

if(index < sz)
{
dest[dest_index] = src[index];
}
}