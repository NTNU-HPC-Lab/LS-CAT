#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void skip_res_add(size_t sz, float_t* f5, float* f1, float_t* skip_out_sum, size_t stride)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

if(index < sz)
{
skip_out_sum[index] += f5[index+stride];
f1[index] += f5[index];
}
}