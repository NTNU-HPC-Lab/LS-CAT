#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void affine_transform(size_t sz, float_t* audio, float_t* end_out, size_t stride)
{
size_t index = blockDim.x * blockIdx.x + threadIdx.x;

if(index < sz)
{
audio[index+stride] = (audio[index+stride]-end_out[index])/expf(end_out[index+stride]);
}
}