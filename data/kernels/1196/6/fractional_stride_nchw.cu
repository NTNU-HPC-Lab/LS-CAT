#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fractional_stride_nchw(size_t num_values, size_t stride, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
size_t index = blockIdx.x*blockDim.x + threadIdx.x;
if(index < num_values)
{
size_t frame_id = (index/ ld_src)*ld_dest + (stride)*(index%ld_src) ;
dest[frame_id] = src[index];
}
}