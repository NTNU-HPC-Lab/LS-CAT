#include "hip/hip_runtime.h"
#include "includes.h"
/* Vector addition deom on GPU

To compile: nvcc -o testprog1 testprog1.cu

*/

using namespace std;


#define FIRST_RUN 0
// Boundaries in physical units on the lens plane
const float WL  = 10.0;
const float XL1 = -WL;
const float XL2 =  WL;
const float YL1 = -WL;
const float YL2 =  WL;

// Source star parameters. You can adjust these if you like - it is
// interesting to look at the different lens images that result
const float rsrc = 0.1;      // radius
const float ldc  = 0.5;      // limb darkening coefficient
const float xsrc = 0.0;      // x and y centre on the map
const float ysrc = 0.0;

// Used to time code. OK for single threaded programs but not for
// multithreaded programs. See other demos for hints at timing CUDA
// code.
__global__ void ray_shoot(int *maxX, int *maxY, float *lens_scale, float *xlens, float *ylens, float*eps, int *num_lenses, float *dev_arr)
{
int threadBlockPos = (blockIdx.x * blockDim.x) + threadIdx.x;

int y = threadBlockPos / (*maxY);
int x = threadBlockPos - ((*maxX) * y);

const float rsrc2 = rsrc * rsrc;

float xl = XL1 + x * (*lens_scale);
float yl = YL1 + y * (*lens_scale);
float xs;
float ys;

float dx, dy, dr;

xs = xl;
ys = yl;

for(int p = 0; p < (*num_lenses); ++p){
dx = xl - xlens[p];
dy = yl - ylens[p];
dr = dx * dx + dy * dy;
xs -= eps[p] * dx / dr;
ys -= eps[p] * dy / dr;
}

float xd = xs - xsrc;
float yd = ys - ysrc;
float sep2 = (xd * xd) + (yd * yd);

if(sep2 < rsrc2){
float mu = sqrtf(1.0f-sep2/rsrc2);
dev_arr[threadBlockPos] = 1.0 - ldc * (1-mu);
}
}