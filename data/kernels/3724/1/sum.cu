#include "hip/hip_runtime.h"
#include "includes.h"
/*
152096 - William Matheus
Friendly Numbers
Programacao Paralela e Distribuida
CUDA - 2019/2 - UPF
Programa 2 - Kernel
*/



__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
int i = blockIdx.x * blockDim.x + threadIdx.x + x;
int j;

if (i < size) {
for (j = i + 1; j < size; j++) {
if ((device_num[i] == device_num[j]) && (device_den[i] == device_den[j]))
device_vet[i]++;
}
}
}