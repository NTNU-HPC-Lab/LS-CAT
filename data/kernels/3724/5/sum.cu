#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
int i = blockIdx.x * blockDim.x + threadIdx.x + x;
int j;

if (i < size) {
for (j = i + 1; j < size; j++) {
if ((device_num[i] == device_num[j]) && (device_den[i] == device_den[j]))
device_vet[i]++;
}
}
}