#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
const int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < size) {
float val = input[index];
if (isnan(val) || isinf(val))
*pinned_return = 1;
}
}