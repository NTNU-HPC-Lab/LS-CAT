#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_zero_kernel(float *src, int size)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) src[i] = 0;
}