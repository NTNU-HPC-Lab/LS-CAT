#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeSignature(double* hyperplanes, double* v, int* dimensions, bool* sig, long* hyperp_length) {
long tid = threadIdx.x + blockDim.x * blockIdx.x;

if (tid < *hyperp_length) {
int d_dimensions = *dimensions;
long pos = tid * d_dimensions;
double sum = 0.0;

for (int i = 0; i < d_dimensions; i++)
sum += hyperplanes[i+pos] * v[i];
sig[tid] = (sum>=0);
}
}