#include "hip/hip_runtime.h"
#include "includes.h"
/*
** Projeto de Algoritmos Paralelos
** Multiplicação de Matrizes
*/


#define TAM_BLOCO 16


// Função para rodar na CPU
// Computa R = M * N
//   aM é a altura de M
//   lM é a largura de M
//   lN é a largura de N
__global__ void cuda_multiplicarmatriz(float* M, float* N, float* R, int tamM, int tamN) {

//índice do bloco
int bx = blockIdx.x;
int by = blockIdx.y;

// índice da thread
int tx = threadIdx.x;
int ty = threadIdx.y;

// índice da primeira submatriz de M processado pelo bloco
int mComeco = tamM * TAM_BLOCO * by;

// índice da última submatriz de M processada pelo bloco
int mFim   = mComeco + tamM - 1;

// Tamanho do passo utilizado para interar através das submatrizes de M
int mPasso  = TAM_BLOCO;

// Índice da primeira submatriz de N processada pelo bloco
int nComeco = TAM_BLOCO * bx;

// Tamanho do passo utilizado para interar através das submatrizes de N
int nPasso  = TAM_BLOCO * tamN;

// O elemento computado pela thread
float rRes = 0;

// Varre por todas as submatrizes de M e N requeridas
// para computar o bloco de submatriz
for (int m = mComeco, n = nComeco; m <= mFim; m += mPasso, n += nPasso) {

// Memoria compartilhada para a submatriz de M
__shared__ float Msub[TAM_BLOCO][TAM_BLOCO];

// Memoria compartilhada para a submatriz de N
__shared__ float Nsub[TAM_BLOCO][TAM_BLOCO];

// Carrega as matrizes da memória global para a memória
// compartilhada. Cada thread carreg um elemento de cada
// matriz
Msub[ty][tx] = M[m + tamM * ty + tx];
Nsub[ty][tx] = N[n + tamN * ty + tx];

// Sincroniza para garantir que todas as matrizes foram
// carregadas
__syncthreads();

// Multiplica as duas matrizes.
// Cada thread computa um elemento
// do bloco da submatriz
for (int i = 0; i < TAM_BLOCO; ++i)
rRes += Msub[ty][i] * Nsub[i][tx];

// Sincroniza para grantir que a computação de multiplicação
// está feita antes de carregar duas novas submatrizes de
// M e N na próxima interação
__syncthreads();
}
// Esscre o bloco da sumatriz na memória global
// Cada thread escreve  um único elemento
int r = tamN * TAM_BLOCO * by + TAM_BLOCO * bx;
R[r + tamN * ty + tx] = rRes;
}