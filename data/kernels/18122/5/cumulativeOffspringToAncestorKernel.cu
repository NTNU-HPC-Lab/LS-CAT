#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cumulativeOffspringToAncestorKernel(const int* cumulativeOffspring, int* ancestor, int numParticles) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx >= numParticles || idx < 0) return;

int start = idx == 0 ? 0 : cumulativeOffspring[idx - 1];
int numCurrentOffspring = cumulativeOffspring[idx] - start;
for(int j = 0; j < numCurrentOffspring; j++)
ancestor[start+j] = idx;
}