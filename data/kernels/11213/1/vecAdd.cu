#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* A, float* B, float* C) {
//threadIdx.x is a build-in variable provided by CUDA runtime
int i = threadIdx.x;
A[i] = 0;
B[i] = 0;
C[i] = A[i] + B[i];
}