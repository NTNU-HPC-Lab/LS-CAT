#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAddInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < cnt) {
out[tid] = in0[tid] + in1[tid];
}
}