#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
float h = hsv.x;
float s = hsv.y;
float v = hsv.z;

float r, g, b;
float f, p, q, t;

if (s == 0) {
r = g = b = v;
} else {
int index = (int) floorf(h);
f = h - index;
p = v*(1-s);
q = v*(1-s*f);
t = v*(1-s*(1-f));
if(index == 0){
r = v; g = t; b = p;
} else if(index == 1){
r = q; g = v; b = p;
} else if(index == 2){
r = p; g = v; b = t;
} else if(index == 3){
r = p; g = q; b = v;
} else if(index == 4){
r = t; g = p; b = v;
} else {
r = v; g = p; b = q;
}
}
r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
b = (b < 0) ? 0 : ((b > 1) ? 1 : b);
return make_float3(r, g, b);
}
__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
float r = rgb.x;
float g = rgb.y;
float b = rgb.z;

float h, s, v;
float max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
float min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
float delta = max - min;
v = max;
if(max == 0){
s = 0;
h = -1;
}else{
s = delta/max;
if(r == max){
h = (g - b) / delta;
} else if (g == max) {
h = 2 + (b - r) / delta;
} else {
h = 4 + (r - g) / delta;
}
if (h < 0) h += 6;
}
return make_float3(h, s, v);
}
__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
int size = batch * w * h;
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(id >= size) return;
int x = id % w;
id /= w;
int y = id % h;
id /= h;
float rshift = rand[0];
float gshift = rand[1];
float bshift = rand[2];
float r0 = rand[8*id + 0];
float r1 = rand[8*id + 1];
float r2 = rand[8*id + 2];
float r3 = rand[8*id + 3];

saturation = r0*(saturation - 1) + 1;
saturation = (r1 > .5f) ? 1.f/saturation : saturation;
exposure = r2*(exposure - 1) + 1;
exposure = (r3 > .5f) ? 1.f/exposure : exposure;

size_t offset = id * h * w * 3;
image += offset;
float r = image[x + w*(y + h*0)];
float g = image[x + w*(y + h*1)];
float b = image[x + w*(y + h*2)];
float3 rgb = make_float3(r,g,b);
if(train){
float3 hsv = rgb_to_hsv_kernel(rgb);
hsv.y *= saturation;
hsv.z *= exposure;
rgb = hsv_to_rgb_kernel(hsv);
} else {
shift = 0;
}
image[x + w*(y + h*0)] = rgb.x*scale + translate + (rshift - .5f)*shift;
image[x + w*(y + h*1)] = rgb.y*scale + translate + (gshift - .5f)*shift;
image[x + w*(y + h*2)] = rgb.z*scale + translate + (bshift - .5f)*shift;
}