#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "computePointHessian1.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *x = NULL;
hipMalloc(&x, XSIZE*YSIZE);
float *y = NULL;
hipMalloc(&y, XSIZE*YSIZE);
float *z = NULL;
hipMalloc(&z, XSIZE*YSIZE);
int points_num = 1;
int *valid_points = NULL;
hipMalloc(&valid_points, XSIZE*YSIZE);
int valid_points_num = 1;
double *dh_ang = NULL;
hipMalloc(&dh_ang, XSIZE*YSIZE);
double *ph124 = NULL;
hipMalloc(&ph124, XSIZE*YSIZE);
double *ph134 = NULL;
hipMalloc(&ph134, XSIZE*YSIZE);
double *ph144 = NULL;
hipMalloc(&ph144, XSIZE*YSIZE);
double *ph154 = NULL;
hipMalloc(&ph154, XSIZE*YSIZE);
double *ph125 = NULL;
hipMalloc(&ph125, XSIZE*YSIZE);
double *ph164 = NULL;
hipMalloc(&ph164, XSIZE*YSIZE);
double *ph135 = NULL;
hipMalloc(&ph135, XSIZE*YSIZE);
double *ph174 = NULL;
hipMalloc(&ph174, XSIZE*YSIZE);
double *ph145 = NULL;
hipMalloc(&ph145, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computePointHessian1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph124,ph134,ph144,ph154,ph125,ph164,ph135,ph174,ph145);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
computePointHessian1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph124,ph134,ph144,ph154,ph125,ph164,ph135,ph174,ph145);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
computePointHessian1<<<gridBlock,threadBlock>>>(x,y,z,points_num,valid_points,valid_points_num,dh_ang,ph124,ph134,ph144,ph154,ph125,ph164,ph135,ph174,ph145);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}