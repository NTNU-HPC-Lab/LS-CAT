#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeScoreGradientList(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, int voxel_num, double *e_x_cov_x, double *cov_dxd_pi, double gauss_d1, int valid_voxel_num, double *score_gradients)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
int col = blockIdx.y;

if (col < 6) {
double *sg = score_gradients + col * valid_points_num;
double *cov_dxd_pi_mat0 = cov_dxd_pi + col * valid_voxel_num;
double *cov_dxd_pi_mat1 = cov_dxd_pi_mat0 + 6 * valid_voxel_num;
double *cov_dxd_pi_mat2 = cov_dxd_pi_mat1 + 6 * valid_voxel_num;

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];
double d_x = static_cast<double>(trans_x[pid]);
double d_y = static_cast<double>(trans_y[pid]);
double d_z = static_cast<double>(trans_z[pid]);

double tmp_sg = 0.0;

for ( int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
int vid = voxel_id[j];
double tmp_ex = e_x_cov_x[j];

if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
tmp_ex *= gauss_d1;

tmp_sg += ((d_x - centroid_x[vid]) * cov_dxd_pi_mat0[j] + (d_y - centroid_y[vid]) * cov_dxd_pi_mat1[j] + (d_z - centroid_z[vid]) * cov_dxd_pi_mat2[j]) * tmp_ex;
}
}

sg[i] = tmp_sg;
}
}
}