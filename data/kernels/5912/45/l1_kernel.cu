#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
float diff = truth[i] - pred[i];
error[i] = abs(diff);
delta[i] = (diff > 0) ? 1 : -1;
}
}