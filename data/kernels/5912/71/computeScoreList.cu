#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

for (int i = id; i < valid_points_num; i += stride) {

double score_inc = 0;

for (int vid = starting_voxel_id[i]; vid < starting_voxel_id[i + 1]; vid++) {
double tmp_ex = e_x_cov_x[vid];

score_inc += (tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex) ? 0 : -gauss_d1 * tmp_ex;
}

score[i] = score_inc;
}
}