#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lineark(int *ip,int *weight,int *op,int N,int M,int L){
unsigned int input_id = (blockIdx.x*gridDim.y + blockIdx.y + blockIdx.z*gridDim.x*gridDim.y)*blockDim.x + threadIdx.x;
int i = input_id/(M*L);
input_id = input_id%(M*L);
int j = input_id/L;
int k = input_id%L;

int temp = (*(ip + i*L + k))*(*(weight + j*L + k));
atomicAdd((op + i*M + j),temp);
}