#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void padding(int *op,int *ip,int N,int C,int H,int W,int Py,int Px){
unsigned int input_id = (blockIdx.x*gridDim.y + blockIdx.y + blockIdx.z*gridDim.x*gridDim.y)*blockDim.x + threadIdx.x;
int i = input_id/(C*H*W);
input_id = input_id%(C*H*W);
int j = input_id/(H*W);
input_id = input_id%(H*W);
int k = input_id/W;
int l = input_id%W;
*(op + i*C*(H + 2*Py)*(W + 2*Px) + j*(H + 2*Py)*(W + 2*Px) + (k + Py)*(W + 2*Px) + (l + Px)) = *(ip + i*C*H*W + j*H*W + k*W + l);
}