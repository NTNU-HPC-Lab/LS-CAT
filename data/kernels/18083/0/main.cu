#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "conv.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
fmap *input = NULL;
hipMalloc(&input, XSIZE*YSIZE);
int *ip = NULL;
hipMalloc(&ip, XSIZE*YSIZE);
int *weights = NULL;
hipMalloc(&weights, XSIZE*YSIZE);
int R = 1;
int S = 1;
fmap *output = NULL;
hipMalloc(&output, XSIZE*YSIZE);
int Sx = 1;
int Sy = 1;
int *op = NULL;
hipMalloc(&op, XSIZE*YSIZE);
int Px = 1;
int Py = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
conv<<<gridBlock,threadBlock>>>(input,ip,weights,R,S,output,Sx,Sy,op,Px,Py);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
conv<<<gridBlock,threadBlock>>>(input,ip,weights,R,S,output,Sx,Sy,op,Px,Py);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
conv<<<gridBlock,threadBlock>>>(input,ip,weights,R,S,output,Sx,Sy,op,Px,Py);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}