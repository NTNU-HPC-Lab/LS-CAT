#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv(fmap *input,int *ip,int *weights,int R,int S,fmap *output, int Sx, int Sy,int *op,int Px,int Py){
unsigned int input_id = (blockIdx.x*gridDim.y + blockIdx.y + blockIdx.z*gridDim.x*gridDim.y)*blockDim.x + threadIdx.x;
int C,H,W,M,E,F;
//N = input->dim1;
C = input->dim2;
H = input->dim3;
W = input->dim4;
M = output->dim2;
E = output->dim3;
F = output->dim4;
H+=2*Py;
W+=2*Px;
/*unsigned int weight_id = input_id%(C*R*S);
int a = weight_id/(R*S);
weight_id = weight_id%(R*S);
int b = weight_id/S;
int c = weight_id%S;*/
int i = input_id/(M*E*F*C*R*S);
input_id = input_id%(M*E*F*C*R*S);
int j = input_id/(E*F*C*R*S);
input_id = input_id%(E*F*C*R*S);
int k = input_id/(F*C*R*S);
input_id = input_id%(F*C*R*S);
int l = input_id/(C*R*S);
input_id = input_id%(C*R*S);
int m = input_id/(R*S);
input_id = input_id%(R*S);
int n = input_id/S;
int o = input_id%S;

int temp = (*(ip + i*C*H*W + m*H*W + (k*Sy + n)*W + (l*Sx + o)))*(*(weights + j*C*R*S + m*R*S + n*S + o));
atomicAdd((op + i*M*E*F + j*E*F + k*F + l), temp);

/* printf("Input fmap\n");
printf("%d %d %d %d\n",N,C,H,W);
for(int i=0;i<N;i++){
for(int j=0;j<C;j++){
for(int k=0;k<H;k++){
for(int l=0;l<W;l++)
printf("%3d ",ip[i*C*H*W + j*H*W + k*H + l]);
printf("\n");
}
printf("\n\n");
}
printf("\n\n\n");
}

printf("Weight fmap\n");
printf("%d %d %d %d\n",M,C,R,S);
for(int i=0;i<M;i++){
for(int j=0;j<C;j++){
for(int k=0;k<R;k++){
for(int l=0;l<S;l++)
printf("%3d ",weights[i*C*R*S + j*R*S + k*S + l]);
printf("\n");
}
printf("\n\n");
}
printf("\n\n\n");
}*/

}