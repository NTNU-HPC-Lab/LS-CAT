#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RELU(int *ip, int N, int C, int H, int W){
unsigned int input_id = blockDim.x*blockIdx.x + threadIdx.x;
int i = input_id/(C*H*W);
input_id = input_id%(C*H*W);
int j = input_id/(H*W);
input_id = input_id%(H*W);
int k = input_id/(W);
int l = input_id%W;

int temp = *(ip + i*C*H*W + j*H*W + k*W + l);
if(temp<0)
*(ip + i*C*H*W + j*H*W + k*W + l) = 0;

}