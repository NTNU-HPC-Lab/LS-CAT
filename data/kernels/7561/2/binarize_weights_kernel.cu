#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (f >= n) return;
int i = 0;
float mean = 0;
for (i = 0; i < size; ++i) {
mean += fabs(weights[f*size + i]);
}
mean = mean / size;
for (i = 0; i < size; ++i) {
binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
//binary[f*size + i] = weights[f*size + i];
}
}