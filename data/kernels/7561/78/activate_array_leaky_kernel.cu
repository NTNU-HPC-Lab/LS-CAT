#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void activate_array_leaky_kernel(float *x, int n)
{
int index = blockIdx.x*blockDim.x + threadIdx.x;
if (index < n) {
float val = x[index];
x[index] = (val > 0) ? val : val / 10;
}
}