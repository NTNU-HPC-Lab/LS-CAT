#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void binarize_kernel(float *x, int n, float *binary)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (i >= n) return;
binary[i] = (x[i] >= 0) ? 1 : -1;
}