#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if (s >= size) return;
int i = 0;
float mean = 0;
for(i = 0; i < n; ++i){
mean += fabs(input[i*size + s]);
}
mean = mean / n;
for(i = 0; i < n; ++i){
binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
}
}