#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void set_zero_kernel(float *src, int size)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) src[i] = 0;
}