#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int f = i / size;
if (f >= n) return;
float mean = mean_arr_gpu[f];
binary[i] = (weights[i] > 0) ? mean : -mean;
}