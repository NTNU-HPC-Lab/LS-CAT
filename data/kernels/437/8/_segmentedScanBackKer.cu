#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _segmentedScanBackKer(float *maxdist, int *maxdistidx, int *label, float *blockmaxdist, int *blocklabel, int *blockmaxdistidx, int numelements)
{
// 声明共享内存。用来存放中间结果小数组中的元素，也就是输入的原数组的每块最
// 后一个元素。共包含三个信息。
__shared__ float shdcurmaxdist[1];
__shared__ int shdcurlabel[1];
__shared__ int shdcurmaxdistindex[1];


// 状态位，用来标记上一块的最后一个元素的标签值是否和本段第一个元素的标签值
// 相同。
__shared__ int state[1];

// 计算需要进行块间累加位置索引（块外的数组索引）。
int idx = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

// 用每块的第一个线程来读取每块前一块的最后一个元素，从中间结果数组中读取。
if (threadIdx.x == 0) {
shdcurmaxdist[0] = blockmaxdist[blockIdx.x];
shdcurlabel[0] = blocklabel[blockIdx.x];
shdcurmaxdistindex[0] = blockmaxdistidx[blockIdx.x];
// 用 state 来记录上一块的最后一个元素的标签值是否和本段第一个元素的
// 标签值相同，相同则为 1，不同则为 0。
state[0] = (label[idx] == shdcurlabel[0]);
}

// 块内同步。
__syncthreads();

// 如果状态位为 0，说明上一块和本块无关，不在一个区域内，直接返回。
if (state[0] == 0)
return;
// 如果数组索引大于数组长度，直接返回。
if (idx >= numelements)
return;
// 如果当前位置处的标签值和目前已知的最大垂距的标签值相同，并且垂距小于目前
// 已知的最大垂距，那么更新当前位置处的最大垂距记录和最大垂距位置的索引。
if (label[idx] == shdcurlabel[0] && maxdist[idx] < shdcurmaxdist[0]) {
maxdist[idx] = shdcurmaxdist[0];
maxdistidx[idx] = shdcurmaxdistindex[0];
}
}