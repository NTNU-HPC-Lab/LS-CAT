#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_check(int threads, uint64_t *data, uint32_t *results, uint64_t target)
{
__shared__ uint32_t tmp[512/32];

int thread = (blockDim.x * blockIdx.x + threadIdx.x);

if(threadIdx.x < (512/32))
tmp[threadIdx.x] = 0;

__syncthreads();

if (thread < threads)
{
uint64_t highword = data[threads*3 + thread];
if(highword < target){
atomicOr(&tmp[threadIdx.x/32], 1 << (threadIdx.x%32));
}

__syncthreads();
if(threadIdx.x < (512/32))
results[blockIdx.x*(4096/32) + threadIdx.x] = tmp[threadIdx.x];
}
}