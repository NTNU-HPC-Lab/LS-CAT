#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _ele_add(float *m, float *target, float val, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
target[tid] = val + m[tid];
}
}