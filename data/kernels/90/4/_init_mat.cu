#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _init_mat(float *m, float val, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
m[tid] = val;
}
}