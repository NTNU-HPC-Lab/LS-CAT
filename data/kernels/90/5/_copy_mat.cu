#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _copy_mat(float *m, float* target, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
target[tid] = m[tid];
}
}