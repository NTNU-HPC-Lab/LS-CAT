#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_pooling_kernel(float *feature_map, float *probs, float *target, int feature_map_size, int feature_map_num, int pooling_rate, float *rnd_array, int rnd_num){
__shared__ float shFm[16*MAX_POOLING_RATE][16*MAX_POOLING_RATE];

int imgIdx = blockIdx.y / (feature_map_size / 16 / pooling_rate);
int fmIdx = blockIdx.x / (feature_map_size / 16 / pooling_rate);
int tx = (blockIdx.x % (feature_map_size / pooling_rate / 16)) * 16 + threadIdx.x;
int ty = (blockIdx.y % (feature_map_size / pooling_rate / 16)) * 16 + threadIdx.y;
int subsample_size = feature_map_size / pooling_rate;

int rnd_index = ((blockIdx.y * blockDim.y + threadIdx.y) * (blockIdx.x * blockDim.x)  + threadIdx.x ) % rnd_num;
float rnd = rnd_array[rnd_index];

float *fm = feature_map + imgIdx * feature_map_num * feature_map_size * feature_map_size +
fmIdx * feature_map_size * feature_map_size;

probs = probs + imgIdx * feature_map_num * feature_map_size * feature_map_size +
fmIdx * feature_map_size * feature_map_size;

target = target + imgIdx * feature_map_num * subsample_size * subsample_size +
fmIdx * subsample_size * subsample_size;

for(int i = 0; i < pooling_rate; i++){
for(int j = 0; j < pooling_rate; j++){
shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j] =
fm[(ty*pooling_rate+i) * feature_map_size + (tx*pooling_rate+j)];
}
}

__syncthreads();

float sum = 0;
for(int i = 0; i < pooling_rate; i++){
for(int j = 0; j < pooling_rate; j++){
if(shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j] > 50){
shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j] = 50.0f;
}
shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j] =
__expf(shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j]);
sum += shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j];
}
}
for(int i = 0; i < pooling_rate; i++){
for(int j = 0; j < pooling_rate; j++){
shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j] =
__fdividef(shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j], (1.0f + sum));
probs[(ty*pooling_rate+i) * feature_map_size + (tx*pooling_rate+j)] =
shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j];
fm[(ty*pooling_rate+i) * feature_map_size + (tx*pooling_rate+j)] = 0;
}
}

sum = 0;
bool isStop = false;
for(int i = 0; i < pooling_rate && !isStop; i++){
for(int j = 0; j < pooling_rate && !isStop; j++){
sum += shFm[threadIdx.y*pooling_rate+i][threadIdx.x*pooling_rate+j];
if(rnd < sum){
fm[(ty*pooling_rate+i) * feature_map_size + (tx*pooling_rate+j)] = 1;
isStop = true;
}
}
}
if(isStop){
target[threadIdx.y*subsample_size+threadIdx.x] = 1;
}else{
target[threadIdx.y*subsample_size+threadIdx.x] = 0;
}
}