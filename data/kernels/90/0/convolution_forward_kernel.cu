#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_forward_kernel(float *input, float *filters, float *feature_map, float *hbias, int input_size, int channel_num, int feature_map_size, int filter_size, int filter_num, int lu_padding, float sigma){
__shared__ float shImg[32+MAX_FILETER_SIZE-1][32+MAX_FILETER_SIZE-1];
__shared__ float shFilter[MAX_FILETER_SIZE][MAX_FILETER_SIZE];

int imgIdx = blockIdx.y / (input_size / 32);
int filterIdx = blockIdx.x / (input_size / 32);
int tx = blockIdx.x % (input_size / 32) * 32 + threadIdx.x;
int ty = blockIdx.y % (input_size / 32) * 32 + threadIdx.y;

float *target = feature_map +
imgIdx * feature_map_size * feature_map_size * filter_num +
feature_map_size * feature_map_size * filterIdx +
ty * feature_map_size + tx;

float local_target = 0.0f;

for(int g = 0; g < channel_num; g++){

if(threadIdx.x < filter_size && threadIdx.y < filter_size){
shFilter[threadIdx.y][threadIdx.x] =
filters[filterIdx * channel_num * filter_size * filter_size +
+ g * filter_size * filter_size +
threadIdx.y * filter_size + threadIdx.x];
}
__syncthreads();

float *img = input + imgIdx * input_size * input_size * channel_num
+ g * input_size * input_size;

float *shImgLoad = &shImg[threadIdx.y][threadIdx.x];
if(tx < lu_padding || ty < lu_padding){
*shImgLoad = 0;
}else{
*shImgLoad = img[(ty-lu_padding) * input_size + (tx-lu_padding)];
}

if(threadIdx.x < MAX_FILETER_SIZE-1){
shImgLoad = &shImg[threadIdx.y][threadIdx.x+32];
if(ty < lu_padding || (tx+32) >= (input_size+lu_padding)){
*shImgLoad = 0;
}else{
*shImgLoad = img[(ty-lu_padding) * input_size +
(tx+32-lu_padding)];
}
}

if(threadIdx.y < MAX_FILETER_SIZE-1){
shImgLoad = &shImg[threadIdx.y+32][threadIdx.x];
if(tx < lu_padding || (ty+32) >= (input_size+lu_padding)){
*shImgLoad = 0;
}else{
*shImgLoad = img[(ty+32-lu_padding) * input_size +
(tx-lu_padding)];
}

if(threadIdx.x < MAX_FILETER_SIZE-1){
shImgLoad = &shImg[threadIdx.y+32][threadIdx.x+32];
if((ty+32) >= (input_size+lu_padding) ||
(tx+32) >= (input_size+lu_padding)){
*shImgLoad = 0;
}else{
*shImgLoad = img[(ty+32-lu_padding) * input_size +
(tx+32-lu_padding)];
}
}
}
__syncthreads();

float *imgPtr = &shImg[threadIdx.y][threadIdx.x];

for(int i = 0; i < filter_size; i++){
for(int j = 0; j < filter_size; j++){
local_target += imgPtr[j] * shFilter[i][j];
}
imgPtr += 32 + MAX_FILETER_SIZE - 1;
}

__syncthreads();

}

local_target += hbias[filterIdx];
local_target *= __fdividef(1.0f , sigma * sigma);
*target = local_target;

}