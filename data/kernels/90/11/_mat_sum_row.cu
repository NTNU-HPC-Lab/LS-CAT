#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_sum_row(float *m, float *target,int nrow, int ncol){
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if(tid < nrow){
float sum = 0;
for(int i = 0; i < ncol; i++){
sum += m[tid*ncol+i];
}
target[tid] = sum;
}
}