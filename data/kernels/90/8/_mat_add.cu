#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_add(float *ma, float *mb, float *target, float sa, float sb, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
target[tid] = sa * ma[tid] + sb * mb[tid];
}
}