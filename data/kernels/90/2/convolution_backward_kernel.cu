#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_backward_kernel(float *y_h, float *filters, float *vbias, float *target, float *y_v, int input_size, int lu_padding, int channel_num, int feature_map_size, int filter_num, int filter_size, float *rnd_array, int rnd_num){
int imgIdx = blockIdx.y / (input_size / 16);
int channelIdx = blockIdx.x / (input_size / 16);
int tx = (blockIdx.x % (input_size / 16)) * 16 + threadIdx.x;
int ty = (blockIdx.y % (input_size / 16)) * 16 + threadIdx.y;
int padding = (filter_size - 1);

int rnd_index = ((blockIdx.y * blockDim.y + threadIdx.y) * (blockIdx.x * blockDim.x)  + threadIdx.x ) % rnd_num;
float rnd = rnd_array[rnd_index];

__shared__ float shHidden[16+2*(MAX_FILETER_SIZE-1)][16+2*(MAX_FILETER_SIZE-1)];
__shared__ float shFlipFilter[MAX_FILETER_SIZE][MAX_FILETER_SIZE];
float local_target = 0.0f;

target = target + imgIdx * channel_num * input_size * input_size +
channelIdx * input_size * input_size;

float *target_y_v = y_v + imgIdx * channel_num * input_size * input_size +
channelIdx * input_size * input_size;

__syncthreads();


for(int f = 0; f < filter_num; f++){
float *cur_y_h = y_h + imgIdx * filter_num * feature_map_size * feature_map_size +
f * feature_map_size * feature_map_size;

float *cur_filter = filters + f * channel_num * filter_size * filter_size +
channelIdx * filter_size * filter_size;

if(threadIdx.x < filter_size && threadIdx.y < filter_size){
shFlipFilter[threadIdx.y][threadIdx.x] =
cur_filter[(filter_size-1-threadIdx.y)*filter_size + filter_size-1-threadIdx.x];
}

float *shHiddenLoad = &shHidden[threadIdx.y][threadIdx.x];
if(tx < padding || ty < padding){
*shHiddenLoad = 0;
}else{
*shHiddenLoad = cur_y_h[(ty-padding) * input_size +
(tx-padding)];
}

if(threadIdx.x < 2 * padding){
shHiddenLoad = &shHidden[threadIdx.y][threadIdx.x+16];
if(ty < padding || (tx+16) >= (feature_map_size+padding)){
*shHiddenLoad = 0;
}else{
*shHiddenLoad = cur_y_h[(ty-padding) * feature_map_size +
(tx+16-padding)];
}
}

if(threadIdx.y < 2 * padding){
shHiddenLoad = &shHidden[threadIdx.y+16][threadIdx.x];
if(tx < padding || (ty+16) >= (feature_map_size+padding)){
*shHiddenLoad = 0;
}else{
*shHiddenLoad = cur_y_h[(ty+16-padding) * feature_map_size +
(tx-padding)];
}

if(threadIdx.x < 2 * padding){
shHiddenLoad = &shHidden[threadIdx.y+16][threadIdx.x+16];
if((ty+16) >= (feature_map_size+padding) ||
(tx+16) >= (feature_map_size+padding)){
*shHiddenLoad = 0;
}else{
*shHiddenLoad = cur_y_h[(ty+16-padding) * feature_map_size +
(tx+16-padding)];
}
}
}

__syncthreads();

for(int i = 0; i < filter_size; i++){
for(int j = 0; j < filter_size; j++){
target[ty*input_size+tx] +=
local_target +=
shHidden[threadIdx.y+i+lu_padding][threadIdx.x+j+lu_padding] *
shFlipFilter[i][j];
}
}

__syncthreads();
}
local_target += vbias[channelIdx];
//local_target = expf(-local_target);
//local_target = __fdividef(1.0f , (1.0f + local_target));
if(rnd < local_target){
target_y_v[ty*input_size+tx] = 1;
}else{
target_y_v[ty*input_size+tx] = 0;
}
target[ty*input_size+tx] = local_target;
}