#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_mul(float *ma, float *mb, float *target, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
target[tid] = ma[tid] * mb[tid];
}
}