#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_sum_col(float *m, float *target,int nrow, int ncol){
int tid = blockIdx.x * blockDim.x + threadIdx.x;

if(tid < ncol){
float sum = 0;
for(int i = 0; i < nrow; i++){
sum += m[i*ncol+tid];
}
target[tid] = sum;
}
}