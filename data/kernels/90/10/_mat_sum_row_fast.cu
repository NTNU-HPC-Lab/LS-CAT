#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_sum_row_fast(float *m, float *target,int nrow, int ncol, int agg_col){
int tx = blockIdx.x * blockDim.x + threadIdx.x;

__shared__ float accum[NUM_THREAD_PER_ROW];

if(tx < ncol){
accum[threadIdx.x] = m[blockIdx.y*ncol+tx];
}else{
accum[threadIdx.x] = 0.0f;
}
__syncthreads();

if(NUM_THREAD_PER_ROW >= 512){
if(threadIdx.x < 256)
accum[threadIdx.x] += accum[threadIdx.x+256];
__syncthreads();
}

if(NUM_THREAD_PER_ROW >= 256){
if(threadIdx.x < 128)
accum[threadIdx.x] += accum[threadIdx.x+128];
__syncthreads();
}

//NUM_THREAD_PER_ROW at least 128
if(threadIdx.x < 64)
accum[threadIdx.x] += accum[threadIdx.x+64];
__syncthreads();

if(threadIdx.x < 32){
accum[threadIdx.x] += accum[threadIdx.x+32];
accum[threadIdx.x] += accum[threadIdx.x+16];
accum[threadIdx.x] += accum[threadIdx.x+8];
accum[threadIdx.x] += accum[threadIdx.x+4];
accum[threadIdx.x] += accum[threadIdx.x+2];
accum[threadIdx.x] += accum[threadIdx.x+1];
}
target[blockIdx.y*agg_col+blockIdx.x] = accum[0];
}