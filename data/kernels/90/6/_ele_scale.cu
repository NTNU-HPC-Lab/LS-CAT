#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _ele_scale(float *m, float *target, float scaler, int len){
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < len){
target[tid] = scaler * m[tid];
}
}