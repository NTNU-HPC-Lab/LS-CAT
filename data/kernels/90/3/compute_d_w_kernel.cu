#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_d_w_kernel(float *v, float *h, float *dw, bool is_init, int input_size, int lu_padding, int channel_num, int filter_num, int filter_size, int feature_map_size){

int imgIdx = blockIdx.y / (feature_map_size / 32);
int filterIdx = blockIdx.x / (channel_num * feature_map_size / 32);
int channelIdx = (blockIdx.x % (channel_num * feature_map_size / 32)) /
(feature_map_size / 32);
int tx = (blockIdx.x % (channel_num * feature_map_size / 32)) %
(feature_map_size / 32) *32 + threadIdx.x;
int ty = (blockIdx.y % (feature_map_size / 32)) * 32 + threadIdx.y;

__shared__ float shV[32+MAX_FILETER_SIZE][32+MAX_FILETER_SIZE];
__shared__ float shH[32][32];

float sign;
if(is_init){
sign = 1.0f;
}else{
sign = -1.0f;
}

v = v + imgIdx * channel_num * input_size * input_size +
channelIdx * input_size * input_size;

h = h + imgIdx * filter_num * feature_map_size * feature_map_size +
filterIdx * feature_map_size * feature_map_size;

dw = dw + filterIdx * channel_num * filter_size * filter_size +
channelIdx * filter_size * filter_size;

float local_dw = 0.0f;

for(int loadX = 0; loadX <= 32; loadX += filter_size){
for(int loadY = 0; loadY <= 32; loadY += filter_size){
if(loadX < 32 && loadY < 32){
//TODO:feature map overflow
shH[threadIdx.y+loadY][threadIdx.x+loadX] =
h[(ty+loadY)*feature_map_size + (tx+loadX)];
}
if((tx+loadX) < lu_padding ||
(ty+loadY) < lu_padding ||
(tx+loadX) >= (input_size+lu_padding) ||
(ty+loadY) >= (input_size+lu_padding)){
shV[threadIdx.y+loadY][threadIdx.x+loadX] = 0;
}else{
shV[threadIdx.y+loadY][threadIdx.x+loadX] =
v[(ty+loadY-lu_padding)*input_size + (tx+loadX-lu_padding)];
}
}
}

__syncthreads();

for(int i = 0; i < 32; i++){
for(int j = 0; j < 32; j++){
local_dw += shV[threadIdx.y+i][threadIdx.x+j] *
shH[i][j];
}
}

atomicAdd(dw + threadIdx.y*filter_size + threadIdx.x, sign * local_dw);
}