#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute(int *v1,int *v2, int *v3, int N){
//blockIdx.x (0-2) threadIdx.x (0-99)
if(blockIdx.x==2){
v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x-2)*N)+threadIdx.x]*v2[((blockIdx.x-1)*N)+threadIdx.x] -
v1[((blockIdx.x-1)*N)+threadIdx.x]*v2[((blockIdx.x-2)*N)+threadIdx.x];
}else if(blockIdx.x==1){
v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x+1)*N)+threadIdx.x]*v2[((blockIdx.x-1)*N)+threadIdx.x] -
v1[(N*(blockIdx.x-1))+threadIdx.x]*v2[((blockIdx.x+1)*N)+threadIdx.x];
}else{
v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x+1)*N)+threadIdx.x]*v2[((blockIdx.x+2)*N)+threadIdx.x] -
v2[((blockIdx.x+1)*N)+threadIdx.x]*v1[((blockIdx.x+2)*N)+threadIdx.x];
}
}