#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(float *A, float *B, float *C, int n)
{
// Each thread computes a single element of C
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

float sum = 0;
for (int i = 0; i < n; ++i) {
sum += (A[row*n + i] * B[i*n + col]);
}

C[row*n + col] = sum;
printf("\n Block[%d][%d] : Thread[%d][%d] : Product = %.2f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, sum);
}