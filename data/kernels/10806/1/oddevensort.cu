#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void oddevensort(int *in, int *out, int size)
{
bool oddeven=true;
__shared__ bool swappedodd;
__shared__ bool swappedeven;
int temp;
swappedodd=true;
swappedeven=true;

while(true)
{
if(oddeven==true)
{
printf(" \n Swapping at odd locations ");
__syncthreads();
swappedodd=false;
__syncthreads();

int idx=threadIdx.x + blockIdx.x * blockDim.x;
if(idx < (size / 2))
{
if(in[2 * idx] > in[2 * idx +1])
{
printf("\n Thread Id %d : is swapping %d <-> %d  \n Thread Id %d : [%d] <-> [%d] \n ", idx, in[2 * idx] ,  in[2 * idx + 1], idx, 2 * idx, (2 * idx +1));

temp = in[2 * idx];
in [2 * idx]= in[2 * idx + 1];
in [2 * idx + 1]=temp;
swappedodd = true;
}
}

__syncthreads();
}

else
{
//printf("Swapping at even locations \n ");
__syncthreads();
swappedeven=false;
__syncthreads();

int idx=threadIdx.x + blockIdx.x * blockDim.x;
if(idx < (size / 2) - 1)
{
if(in[2 * idx + 1] > in[2 * idx +2])
{
printf("\n Thread Id %d : is swapping %d <-> %d  \n Thread Id %d : [%d] <-> [%d] \n ", idx, in[2 * idx + 1] ,  in[2 * idx + 2], idx, 2 * idx + 1, (2 * idx +2));

temp = in[2 * idx + 1];
in [2 * idx + 1]= in[2 * idx + 2];
in [2 * idx + 2] = temp;
swappedeven=true;
}
}
__syncthreads();
}

if(!(swappedodd || swappedeven ))
break;
oddeven = !oddeven;
}

__syncthreads();

int idx =threadIdx.x;

if(idx < size)
out[idx] = in[idx];
}