#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_pillar_feature_kernel( float* dev_pillar_point_feature_in_coors, float* dev_pillar_point_feature, float* dev_pillar_coors, int* dev_x_coors, int* dev_y_coors, float* dev_num_points_per_pillar, const int max_points, const int num_point_feature, const int grid_x_size) {
int ith_pillar = blockIdx.x;
int num_points_at_this_pillar = dev_num_points_per_pillar[ith_pillar];
int ith_point = threadIdx.x;
if (ith_point >= num_points_at_this_pillar) {
return;
}
int x_ind = dev_x_coors[ith_pillar];
int y_ind = dev_y_coors[ith_pillar];
int pillar_ind = ith_pillar * max_points * num_point_feature +
ith_point * num_point_feature;
int coors_ind = y_ind * grid_x_size * max_points * num_point_feature +
x_ind * max_points * num_point_feature +
ith_point * num_point_feature;
for (int i = 0; i < num_point_feature; ++i) {
dev_pillar_point_feature[pillar_ind + i] =
dev_pillar_point_feature_in_coors[coors_ind + i];
}

float coor_x = static_cast<float>(x_ind);
float coor_y = static_cast<float>(y_ind);
dev_pillar_coors[ith_pillar * 4 + 0] = 0;  // batch idx
dev_pillar_coors[ith_pillar * 4 + 1] = 0;  // z
dev_pillar_coors[ith_pillar * 4 + 2] = coor_y;
dev_pillar_coors[ith_pillar * 4 + 3] = coor_x;
}