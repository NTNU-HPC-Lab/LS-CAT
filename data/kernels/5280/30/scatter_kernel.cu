#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int max_num_pillars_, const int grid_x_size, const int grid_y_size) {
int i_pillar = blockIdx.x;
int i_feature = threadIdx.x;
int x_ind = x_coors[i_pillar];
int y_ind = y_coors[i_pillar];
float feature = pfe_output[i_feature * max_num_pillars_ + i_pillar];
scattered_feature[i_feature * grid_y_size * grid_x_size +
y_ind * grid_x_size + x_ind] = feature;
}