#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_kernel(const int n, const float alpha, float *y) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);
i += blockDim.x * gridDim.x) {
y[i] = alpha;
}
}