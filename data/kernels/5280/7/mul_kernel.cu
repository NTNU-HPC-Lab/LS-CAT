#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(const int n, const float *a, const float *b, float *y) {
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);
i += blockDim.x * gridDim.x) {
y[i] = a[i] * b[i];
}
}