#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_lower_left_gpu(int *iRow, int *jCol, unsigned int *rind_L, unsigned int *cind_L, const int nnz_L) {
int i = threadIdx.x;

if (i < nnz_L) {
iRow[i] = rind_L[i];
jCol[i] = cind_L[i];
}
}