#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multi_scale_kernel(const float *data_in, const float *scale, float *data_out, int width, int height) {
const int x = blockDim.x * blockIdx.x + threadIdx.x;
const int y = blockDim.y * blockIdx.y + threadIdx.y;

if (x < width && y < height) {
int index = y * width + x;
data_out[index] = data_in[index] * scale[y];
}
}