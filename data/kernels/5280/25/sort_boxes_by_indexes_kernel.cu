#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sort_boxes_by_indexes_kernel( float* filtered_box, int* filtered_label, int* filtered_dir, float* box_for_nms, int* indexes, int filter_count, float* sorted_filtered_boxes, int* sorted_filtered_label, int* sorted_filtered_dir, float* sorted_box_for_nms, const int num_box_corners, const int num_output_box_feature) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < filter_count) {
int sort_index = indexes[tid];
sorted_filtered_boxes[tid * num_output_box_feature + 0] =
filtered_box[sort_index * num_output_box_feature + 0];
sorted_filtered_boxes[tid * num_output_box_feature + 1] =
filtered_box[sort_index * num_output_box_feature + 1];
sorted_filtered_boxes[tid * num_output_box_feature + 2] =
filtered_box[sort_index * num_output_box_feature + 2];
sorted_filtered_boxes[tid * num_output_box_feature + 3] =
filtered_box[sort_index * num_output_box_feature + 3];
sorted_filtered_boxes[tid * num_output_box_feature + 4] =
filtered_box[sort_index * num_output_box_feature + 4];
sorted_filtered_boxes[tid * num_output_box_feature + 5] =
filtered_box[sort_index * num_output_box_feature + 5];
sorted_filtered_boxes[tid * num_output_box_feature + 6] =
filtered_box[sort_index * num_output_box_feature + 6];

sorted_filtered_label[tid] = filtered_label[sort_index];

sorted_filtered_dir[tid] = filtered_dir[sort_index];

sorted_box_for_nms[tid * num_box_corners + 0] =
box_for_nms[sort_index * num_box_corners + 0];
sorted_box_for_nms[tid * num_box_corners + 1] =
box_for_nms[sort_index * num_box_corners + 1];
sorted_box_for_nms[tid * num_box_corners + 2] =
box_for_nms[sort_index * num_box_corners + 2];
sorted_box_for_nms[tid * num_box_corners + 3] =
box_for_nms[sort_index * num_box_corners + 3];
}
}