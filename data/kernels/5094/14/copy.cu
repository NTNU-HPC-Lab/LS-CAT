#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void copy(int nx,int ny,int nz, Real *in, Real *out) {

int tj = threadIdx.x;
//int td = blockDim.x;

int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
int jj1 =  ((blockIdx.y)*nx*ny + (blockIdx.x)*nx);

out[jj+tj] = in[jj1+tj];
}