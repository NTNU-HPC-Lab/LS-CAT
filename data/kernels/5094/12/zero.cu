#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void zero(int nx, int ny, int nz, Real *z) {

int tj = threadIdx.x;
int td = blockDim.x;

int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

int jj = ((blockIdx.y)*gridDim.x + (blockIdx.x))*blockData;

for (int k=0; k<blockData/td; k++) {
z[jj + tj+ k*td] = 0.0;
}
}