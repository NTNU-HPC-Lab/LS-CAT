#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void partialScan(unsigned int *d_in, unsigned int *d_out, unsigned int *d_total, size_t n)
{
__shared__ unsigned int temp[BLOCK_WIDTH];
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = BLOCK_WIDTH * bx + tx;

if(index < n) {
temp[tx] = d_in[index];
} else { temp[tx] = 0; }
__syncthreads();

// Perform the actual scan
for(int offset = 1; offset < BLOCK_WIDTH; offset <<= 1) {
if(tx + offset < BLOCK_WIDTH) {
temp[tx + offset] += temp[tx];
}
__syncthreads();
}

// Shift when copying the result so as to make it an exclusive scan
if(tx +1 < BLOCK_WIDTH && index + 1 < n) {
d_out[index + 1] = temp[tx];
}
d_out[0] = 0;

// Store the total sum of each block
d_total[bx] = temp[BLOCK_WIDTH - 1];
}