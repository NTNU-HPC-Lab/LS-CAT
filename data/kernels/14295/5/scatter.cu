#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter(unsigned int *d_inVals, unsigned int *d_outVals, unsigned int *d_inPos, unsigned int *d_outPos, unsigned int *d_zerosScan, unsigned int *d_onesScan, unsigned int *d_zerosPredicate, unsigned int *d_onesPredicate, size_t n)
{
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = BLOCK_WIDTH * bx + tx;
int offset = d_zerosScan[n - 1] + d_zerosPredicate[n - 1];

if(index < n) {
int scatterIdx;
if(d_zerosPredicate[index]) {
scatterIdx = d_zerosScan[index];
} else {
scatterIdx = d_onesScan[index] + offset;
}
if(scatterIdx < n) { //sanity check
d_outVals[scatterIdx] = d_inVals[index];
d_outPos[scatterIdx] = d_inPos[index];
}
}
}