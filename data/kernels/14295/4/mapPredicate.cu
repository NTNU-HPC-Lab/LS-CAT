#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapPredicate(unsigned int *d_zeros, unsigned int *d_ones, unsigned int *d_in, unsigned int bit, size_t n)
{
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = BLOCK_WIDTH * bx + tx;

if(index < n) {
unsigned int isOne = (d_in[index] >> bit) & 1;
d_ones[index] = isOne;
d_zeros[index] = 1 - isOne;
}
}