#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapScan(unsigned int *d_array, unsigned int *d_total, size_t n) {
int tx = threadIdx.x;
int bx = blockIdx.x;
int index = BLOCK_WIDTH * bx + tx;

if(index < n) {
d_array[index] += d_total[bx];
}
}