#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RGBToRGBAf(uchar3* srcImage, float4* dstImage, uint32_t width,       uint32_t height)
{
int x, y, pixel;

x = (blockIdx.x * blockDim.x) + threadIdx.x;
y = (blockIdx.y * blockDim.y) + threadIdx.y;

pixel = y * width + x;

if (x >= width)
return;

if (y >= height)
return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);

const float  s  = 1.0f;
const uchar3 px = srcImage[pixel];

dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}