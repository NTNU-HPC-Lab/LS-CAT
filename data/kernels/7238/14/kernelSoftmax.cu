#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void kernelSoftmax( float* x, int channels, float* y)
{

extern __shared__ float mem[];
__shared__ float sum_value;

float number = *(x + blockDim.x*blockIdx.x + threadIdx.x);
float number_exp = __expf(number);

//    sum_value += number_exp ;
/* *
* @TODO: Can do with the help of atomicAdd.
* */
atomicAdd(&sum_value, number_exp);
__syncthreads();

//	mem[threadIdx.x] = number_exp;

/* *
* @TODO: Can do with the help of a for loop. Try different methods and find the time taken.
* */
//	float sum = 0.0f;
//	for (int i=0;i<channels;i++)
//	{
//		sum += mem[i];
//	}

y[blockDim.x*blockIdx.x + threadIdx.x] = __fdiv_rd(number_exp, sum_value);

}