#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void filter_kernel(const float* box_preds, const float* cls_preds, const float* dir_preds, const int* anchor_mask, const float* dev_anchors_px, const float* dev_anchors_py, const float* dev_anchors_pz, const float* dev_anchors_dx, const float* dev_anchors_dy, const float* dev_anchors_dz, const float* dev_anchors_ro, float* filtered_box, float* filtered_score, int* filtered_dir, float* box_for_nms, int* filter_count, const float FLOAT_MIN, const float FLOAT_MAX, const float score_threshold, const int NUM_BOX_CORNERS, const int NUM_OUTPUT_BOX_FEATURE)
{
// boxes ([N, 7] Tensor): normal boxes: x, y, z, w, l, h, r
int tid = threadIdx.x + blockIdx.x * blockDim.x;
//sigmoid funciton
float score = 1/(1+expf(-cls_preds[tid]));
if(anchor_mask[tid] == 1 && score > score_threshold)
{
int counter = atomicAdd(filter_count, 1);
float za = dev_anchors_pz[tid] + dev_anchors_dz[tid]/2;

//decode network output
float diagonal = sqrtf(dev_anchors_dx[tid]*dev_anchors_dx[tid] + dev_anchors_dy[tid]*dev_anchors_dy[tid]);
float box_px = box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 0] * diagonal + dev_anchors_px[tid];
float box_py = box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 1] * diagonal + dev_anchors_py[tid];
float box_pz = box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 2] * dev_anchors_dz[tid] + za;
float box_dx = expf(box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 3]) * dev_anchors_dx[tid];
float box_dy = expf(box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 4]) * dev_anchors_dy[tid];
float box_dz = expf(box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 5]) * dev_anchors_dz[tid];
float box_ro = box_preds[tid*NUM_OUTPUT_BOX_FEATURE + 6] + dev_anchors_ro[tid];

box_pz = box_pz - box_dz/2;

filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 0] = box_px;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 1] = box_py;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 2] = box_pz;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 3] = box_dx;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 4] = box_dy;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 5] = box_dz;
filtered_box[counter*NUM_OUTPUT_BOX_FEATURE + 6] = box_ro;
filtered_score[counter] = score;

int direction_label;
if(dir_preds[tid*2 + 0] < dir_preds[tid*2 + 1])
{
direction_label = 1;
}
else
{
direction_label = 0;
}
filtered_dir[counter] = direction_label;

//convrt normal box(normal boxes: x, y, z, w, l, h, r) to box(xmin, ymin, xmax, ymax) for nms calculation
//First: dx, dy -> box(x0y0, x0y1, x1y0, x1y1)
float corners[NUM_3D_BOX_CORNERS_MACRO] = {float(-0.5*box_dx), float(-0.5*box_dy),
float(-0.5*box_dx), float( 0.5*box_dy),
float( 0.5*box_dx), float( 0.5*box_dy),
float( 0.5*box_dx), float(-0.5*box_dy)};

//Second: Rotate, Offset and convert to point(xmin. ymin, xmax, ymax)
float rotated_corners[NUM_3D_BOX_CORNERS_MACRO];
float offset_corners[NUM_3D_BOX_CORNERS_MACRO];
float sin_yaw = sinf(box_ro);
float cos_yaw = cosf(box_ro);
float xmin = FLOAT_MAX;
float ymin = FLOAT_MAX;
float xmax = FLOAT_MIN;
float ymax = FLOAT_MIN;
for(size_t i = 0; i < NUM_BOX_CORNERS; i++)
{
rotated_corners[i*2 + 0] = cos_yaw*corners[i*2 + 0] - sin_yaw*corners[i*2 + 1];
rotated_corners[i*2 + 1] = sin_yaw*corners[i*2 + 0] + cos_yaw*corners[i*2 + 1];

offset_corners[i*2 + 0] = rotated_corners[i*2 + 0] + box_px;
offset_corners[i*2 + 1] = rotated_corners[i*2 + 1] + box_py;

xmin = fminf(xmin, offset_corners[i*2 + 0]);
ymin = fminf(ymin, offset_corners[i*2 + 1]);
xmax = fmaxf(xmin, offset_corners[i*2 + 0]);
ymax = fmaxf(ymax, offset_corners[i*2 + 1]);
}
// box_for_nms(num_box, 4)
box_for_nms[counter*NUM_BOX_CORNERS + 0] = xmin;
box_for_nms[counter*NUM_BOX_CORNERS + 1] = ymin;
box_for_nms[counter*NUM_BOX_CORNERS + 2] = xmax;
box_for_nms[counter*NUM_BOX_CORNERS + 3] = ymax;

}
}