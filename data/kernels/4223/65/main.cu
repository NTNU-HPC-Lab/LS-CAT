#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "make_pillar_histo_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const float *dev_points = NULL;
hipMalloc(&dev_points, XSIZE*YSIZE);
float *dev_pillar_x_in_coors = NULL;
hipMalloc(&dev_pillar_x_in_coors, XSIZE*YSIZE);
float *dev_pillar_y_in_coors = NULL;
hipMalloc(&dev_pillar_y_in_coors, XSIZE*YSIZE);
float *dev_pillar_z_in_coors = NULL;
hipMalloc(&dev_pillar_z_in_coors, XSIZE*YSIZE);
float *dev_pillar_i_in_coors = NULL;
hipMalloc(&dev_pillar_i_in_coors, XSIZE*YSIZE);
int *pillar_count_histo = NULL;
hipMalloc(&pillar_count_histo, XSIZE*YSIZE);
const int num_points = 1;
const int max_points_per_pillar = 1;
const int GRID_X_SIZE = 1;
const int GRID_Y_SIZE = 1;
const int GRID_Z_SIZE = 1;
const float MIN_X_RANGE = 1;
const float MIN_Y_RANGE = 1;
const float MIN_Z_RANGE = 1;
const float PILLAR_X_SIZE = 1;
const float PILLAR_Y_SIZE = 1;
const float PILLAR_Z_SIZE = 1;
const int NUM_BOX_CORNERS = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
make_pillar_histo_kernel<<<gridBlock,threadBlock>>>(dev_points,dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,pillar_count_histo,num_points,max_points_per_pillar,GRID_X_SIZE,GRID_Y_SIZE,GRID_Z_SIZE,MIN_X_RANGE,MIN_Y_RANGE,MIN_Z_RANGE,PILLAR_X_SIZE,PILLAR_Y_SIZE,PILLAR_Z_SIZE,NUM_BOX_CORNERS);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
make_pillar_histo_kernel<<<gridBlock,threadBlock>>>(dev_points,dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,pillar_count_histo,num_points,max_points_per_pillar,GRID_X_SIZE,GRID_Y_SIZE,GRID_Z_SIZE,MIN_X_RANGE,MIN_Y_RANGE,MIN_Z_RANGE,PILLAR_X_SIZE,PILLAR_Y_SIZE,PILLAR_Z_SIZE,NUM_BOX_CORNERS);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
make_pillar_histo_kernel<<<gridBlock,threadBlock>>>(dev_points,dev_pillar_x_in_coors,dev_pillar_y_in_coors,dev_pillar_z_in_coors,dev_pillar_i_in_coors,pillar_count_histo,num_points,max_points_per_pillar,GRID_X_SIZE,GRID_Y_SIZE,GRID_Z_SIZE,MIN_X_RANGE,MIN_Y_RANGE,MIN_Z_RANGE,PILLAR_X_SIZE,PILLAR_Y_SIZE,PILLAR_Z_SIZE,NUM_BOX_CORNERS);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}