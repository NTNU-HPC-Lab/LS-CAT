#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if(i < n){
float t = truth[i];
float p = pred[i];
error[i] = (t) ? -log(p) : 0;
delta[i] = t-p;
}
}