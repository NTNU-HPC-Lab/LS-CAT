#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointGradients1(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dj_ang, double *pg24, double *pg05, double *pg15, double *pg25)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
__shared__ double j_ang[12];


if (threadIdx.x < 12) {
j_ang[threadIdx.x] = dj_ang[threadIdx.x + 12];
}

__syncthreads();

for (int i = id; i < valid_points_num; i += stride) {
int pid = valid_points[i];

//Orignal coordinates
double o_x = static_cast<double>(x[pid]);
double o_y = static_cast<double>(y[pid]);
double o_z = static_cast<double>(z[pid]);

//Compute point derivatives

pg24[i] = o_x * j_ang[0] + o_y * j_ang[1] + o_z * j_ang[2];
pg05[i] = o_x * j_ang[3] + o_y * j_ang[4] + o_z * j_ang[5];
pg15[i] = o_x * j_ang[6] + o_y * j_ang[7] + o_z * j_ang[8];
pg25[i] = o_x * j_ang[9] + o_y * j_ang[10] + o_z * j_ang[11];
}
}