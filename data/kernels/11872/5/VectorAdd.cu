#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n) {
int i = threadIdx.x;
// no loop for (i = 0; i < n; ++i)
if (i < n)
c[i] = a[i] + b[i];
}