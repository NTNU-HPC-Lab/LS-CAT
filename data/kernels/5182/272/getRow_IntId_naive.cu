#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getRow_IntId_naive(const float * A, int row_id, float * out, int Acols) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < Acols) {
out[id] = A[id + row_id*Acols];
}
}