#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DropoutMaskKernel( float *dropoutMaskPtr, float dropout, int inputSize )
{
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < inputSize)
{
dropoutMaskPtr[i] = dropout > dropoutMaskPtr[i];
/*if (dropoutMaskPtr[i] > dropout)
dropoutMaskPtr[i] = 0.0f;
else
dropoutMaskPtr[i] = 1.0f;*/
}
}