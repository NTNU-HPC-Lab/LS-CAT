#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void crate3Dplot(float* plotValues, float patchSize, int itemsX, int itemsY, float maxValue, float* vertexData)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

int size = itemsX * itemsY;
float texSizeX = 1.0f / itemsX;
float texSizeY = 1.0f / itemsY;

int patchX = threadId / itemsY;
int patchY = itemsY - (threadId % itemsY) - 1;

if (threadId < size)
{
float height = plotValues[threadId] / maxValue;
float gap = 0.000;

float3* vertTop = (float3*)vertexData;
float2* texCoords = (float2*)(vertexData + (60 * size));

//top side
vertTop[threadId * 4].x = patchX * patchSize + gap;
vertTop[threadId * 4].y = height;
vertTop[threadId * 4].z = patchY * patchSize + gap;

texCoords[threadId * 4].x = patchX * texSizeX;
texCoords[threadId * 4].y = patchY * texSizeY;

vertTop[threadId * 4 + 1].x = (patchX + 1) * patchSize - gap;
vertTop[threadId * 4 + 1].y = height;
vertTop[threadId * 4 + 1].z = patchY * patchSize + gap;

texCoords[threadId * 4 + 1].x = (patchX + 1) * texSizeX;
texCoords[threadId * 4 + 1].y = patchY * texSizeY;

vertTop[threadId * 4 + 2].x = (patchX + 1) * patchSize - gap;
vertTop[threadId * 4 + 2].y = height;
vertTop[threadId * 4 + 2].z = (patchY + 1) * patchSize - gap;

texCoords[threadId * 4 + 2].x = (patchX + 1) * texSizeX;
texCoords[threadId * 4 + 2].y = (patchY + 1) * texSizeY;

vertTop[threadId * 4 + 3].x = patchX * patchSize + gap;
vertTop[threadId * 4 + 3].y = height;
vertTop[threadId * 4 + 3].z = (patchY + 1) * patchSize - gap;

texCoords[threadId * 4 + 3].x = patchX * texSizeX;
texCoords[threadId * 4 + 3].y = (patchY + 1) * texSizeY;

float3* vertLeft = (float3*)(vertexData + 12 * size);

//left side
vertLeft[threadId * 4] = vertTop[threadId * 4];
vertLeft[threadId * 4].y = 0;
vertLeft[threadId * 4 + 1] = vertTop[threadId * 4];

vertLeft[threadId * 4 + 2] = vertTop[threadId * 4 + 3];
vertLeft[threadId * 4 + 3] = vertTop[threadId * 4 + 3];
vertLeft[threadId * 4 + 3].y = 0;

float3* vertFar = (float3*)(vertexData + 24 * size);

//far side
vertFar[threadId * 4] = vertTop[threadId * 4 + 2];
vertFar[threadId * 4].y = 0;
vertFar[threadId * 4 + 1] = vertTop[threadId * 4 + 3];
vertFar[threadId * 4 + 1].y = 0;

vertFar[threadId * 4 + 2] = vertTop[threadId * 4 + 3];
vertFar[threadId * 4 + 3] = vertTop[threadId * 4 + 2];

float3* vertNear = (float3*)(vertexData + 36 * size);

//near side
vertNear[threadId * 4] = vertTop[threadId * 4 + 1];
vertNear[threadId * 4].y = 0;
vertNear[threadId * 4 + 1] = vertTop[threadId * 4];
vertNear[threadId * 4 + 1].y = 0;

vertNear[threadId * 4 + 2] = vertTop[threadId * 4];
vertNear[threadId * 4 + 3] = vertTop[threadId * 4 + 1];

float3* vertRight = (float3*)(vertexData + 48 * size);

//right side
vertRight[threadId * 4] = vertTop[threadId * 4 + 2];
vertRight[threadId * 4].y = 0;
vertRight[threadId * 4 + 1] = vertTop[threadId * 4 + 2];

vertRight[threadId * 4 + 2] = vertTop[threadId * 4 + 1];
vertRight[threadId * 4 + 3] = vertTop[threadId * 4 + 1];
vertRight[threadId * 4 + 3].y = 0;
}
}