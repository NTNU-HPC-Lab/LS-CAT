#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbBackgroundKernel(float *target, int inputWidth, int inputHeight, float r, float g, float b)
{
int column = threadIdx.x + blockDim.x * blockIdx.z;
if (column >= inputWidth)
return;

int id = inputWidth * ( blockIdx.y * gridDim.x + blockIdx.x) // blockIdx.x == row, blockIdx.y == color channel
+ column;

int imagePixels = inputWidth * inputHeight;

if (id < 3*imagePixels) // 3 for RGB
{
float color = 0.0f;
switch (blockIdx.y)
{
case 0:
color = r;
break;
case 1:
color = g;
break;
case 2:
color = b;
break;
}
target[id] = color;
}
}