#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearCombinationKernel(float *input1, float input1_coeff, int input1_start_index, float *input2, float input2_coeff, int input2_start_index, float *output, int output_start_index, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if(id < size)
{
output[output_start_index + id] = input1_coeff * input1[input1_start_index + id] + input2_coeff * input2[input2_start_index + id];
}
}