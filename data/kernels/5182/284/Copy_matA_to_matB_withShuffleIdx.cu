#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void Copy_matA_to_matB_withShuffleIdx (float * A , float * B , int size, int cols , float * new_idxs, int max_rows){
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int irow = id / cols;
int icol = id % cols;
if (id<size){
int irow_new = max_rows - 1 - irow; /// it was ascending, so I need to revert it...
int irow_old = new_idxs[irow];
B[irow_new*cols + icol] = A[irow_old*cols + icol];
}
}