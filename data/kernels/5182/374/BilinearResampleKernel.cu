#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size = outputWidth * outputHeight;
float iT, iB;

if (id < size)
{
//output point coordinates
int px = id % outputWidth;
int py = id / outputWidth;

float xRatio = (float)(inputWidth - 1) / (outputWidth - 1);
float yRatio = (float)(inputHeight - 1) / (outputHeight - 1);

//corresponding coordinates in the original image
float x = xRatio * px;
float y = yRatio * py;

//corresponding integer (pixel) coordinates in the original image
int xL = (int)floor(x);
int xR = (int)ceil(x);
int yT = (int)floor(y);
int yB = (int)ceil(y);


//inverse distances to these points
float dL = 1.0f - (x - xL);
float dR = 1.0f - (xR - x);
float dT = 1.0f - (y - yT);
float dB = 1.0f - (yB - y);

//values at those points
float topLeft = input[yT * inputWidth + xL];
float topRight = input[yT * inputWidth + xR];
float bottomLeft = input[yB * inputWidth + xL];
float bottomRight = input[yB * inputWidth + xR];

//linear interpolation in X (i.e., top and bottom pairs of points)
if (xL == xR) { //interpolated points corresponds exactly to one integer x-coordinate in the original image, choose any one of them
iT = topLeft;
iB = bottomLeft;
}
else {
iT = topLeft * dL + topRight * dR;
iB = bottomLeft * dL + bottomRight * dR;
}

//linear interpolation in Y (i.e., linear interpolation of those two points)
if (yT == yB) //interpolated points corresponds exactly to one integer ycoordinate in the original image, choose any one of them
{
output[py * outputWidth + px] = iT;
}
else {
output[py * outputWidth + px] = iT * dT + iB * dB;
}
}
}