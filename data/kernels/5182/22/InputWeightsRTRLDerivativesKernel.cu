#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InputWeightsRTRLDerivativesKernel( float *input, float *hiddenActivationDerivatives, float *recurrentWeights, float *inputWeightRTRLDerivatives, float *previousInputWeightRTRLDerivatives )
{
int partialId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
{
int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
int to = weightId / D_INPUT_UNITS;
int from = weightId % D_INPUT_UNITS;

float sum = 0;
for (int i = 0; i < D_HIDDEN_UNITS; i++)
{
sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousInputWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
}

inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * input[from] + sum);
}
}