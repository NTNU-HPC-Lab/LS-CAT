#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddLocalErrorKernel(  int s1, float *distance, float *localError  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
localError[s1] += distance[s1] * distance[s1];
}

}