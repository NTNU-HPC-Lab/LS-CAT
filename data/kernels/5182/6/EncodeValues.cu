#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void EncodeValuesInternal(float value, float& origin, float& dir, float& output, int squaredMode)
{
if (squaredMode == 1)
{
// origin part:      o * (1 - t)^2
output = (1 - fabs(value)) * (1 - fabs(value)) * origin;
// direction part:   dir * (-t^2 + 2*t)
output += (-value * value + 2 * fabs(value)) * dir;
}
else
{
// origin part:      o * (1 - t)
output = (1 - fabs(value)) * origin;
// direction part:   dir * t
output += fabs(value) * dir;
}
}
__global__  void EncodeValues(float* values, int numOfValues, float* output, int symbolSize, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

__shared__ float s_values[2];

if (threadIdx.x < 2)
{
//clamp to (-1, 1) if square mode is used

if (squaredMode == 1)
{
s_values[threadIdx.x] = fmaxf(fminf(values[threadIdx.x], 1), -1);
}
else
{
s_values[threadIdx.x] = values[threadIdx.x];
}
}

__syncthreads();


if (threadId >= symbolSize)
return;


// X dim
float* dir = (s_values[0] > 0) ? dirX : negDirX;
EncodeValuesInternal(s_values[0], originX[threadId], dir[threadId], output[threadId], squaredMode);

// Y dim
if (numOfValues > 1)
{
dir = (s_values[1] > 0) ? dirY : negDirY;
EncodeValuesInternal(s_values[1], originY[threadId], dir[threadId], output[threadId], squaredMode);
}
}