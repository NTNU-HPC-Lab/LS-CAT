#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void ResetImage(float* im, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size)
im[id] = 0;
}