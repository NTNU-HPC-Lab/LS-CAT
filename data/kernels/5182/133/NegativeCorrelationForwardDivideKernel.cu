#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NegativeCorrelationForwardDivideKernel( float* outputPtr, int thisLayerSize, int inputModelCount )
{
// j: current layer neuron id
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
outputPtr[j] /= (float)inputModelCount;
}
}