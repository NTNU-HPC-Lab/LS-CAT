#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyRectangleCheckBoundsKernel(	float *src, int srcOffset, int srcWidth, int srcHeight, int srcRectX, int srcRectY, int rectWidth, int rectHeight, float *dest, int destOffset, int destWidth, int destRectX, int destRectY, float defaultValue )
{
int id = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

int size = rectWidth * rectHeight;

if (id < size) {

int localX = id % rectWidth;
int localY = id / rectWidth;

int srcPixelX = srcRectX + localX;
int srcPixelY = srcRectY + localY;

int destPixelX = destRectX + localX;
int destPixelY = destRectY + localY;

if (srcPixelX >= 0 && srcPixelX < srcWidth && srcPixelY >= 0 && srcPixelY < srcHeight)
{
(dest + destOffset)[destPixelX + destPixelY * destWidth] = (src + srcOffset)[srcPixelX + srcPixelY * srcWidth];
}
else
{
(dest + destOffset)[destPixelX + destPixelY * destWidth] = defaultValue;
}
}
}