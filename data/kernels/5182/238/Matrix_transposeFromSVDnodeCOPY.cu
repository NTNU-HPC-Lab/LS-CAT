#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_transposeFromSVDnodeCOPY(const float* A, int Acount, int Acols, float* out0)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;

int Arows = Acount/Acols;

int x = id / Arows;
int y = id % Arows;

if (id < Acount)
{
out0[x * Arows + y] = A[y * Acols + x];
}
}