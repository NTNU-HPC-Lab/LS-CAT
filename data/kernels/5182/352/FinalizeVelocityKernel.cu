#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FinalizeVelocityKernel(float* velocities, float* globalFlow, int inputWidth, int inputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size =  inputWidth * inputHeight;

if (id < size)
{
float globalFlowL = sqrtf(globalFlow[0] * globalFlow[0] + globalFlow[1] * globalFlow[1]);
float velocityL = sqrtf(velocities[id] * velocities[id]  + velocities[size + id] * velocities[size + id]);

if (globalFlowL > 0 && velocityL > 0) {

float dot = (globalFlow[0] * velocities[id] + globalFlow[1] * velocities[size + id]) / (globalFlowL * velocityL);

if (dot > 0.7) {
velocities[id] = 0;
velocities[size + id] = 0;
}
}
}
}