#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activateRandomly(float probability, float random)
{
return random < probability;
}
__global__ void RBMRandomActivationKernel( float					*outputPtr, float					*randomPtr, int						size )
{

int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < size)
{
outputPtr[i] = activateRandomly(outputPtr[i], randomPtr[i]);
}
}