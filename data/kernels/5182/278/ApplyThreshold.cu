#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyThreshold( float* probabilitiesInputs, float* binaryOutput, float* probability, int count ) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;

if (id < count)
{
if (probabilitiesInputs[id] < probability[0])
{
binaryOutput[id] = 0.0f;
}
else
{
binaryOutput[id] = 1.0f;
}
}
}