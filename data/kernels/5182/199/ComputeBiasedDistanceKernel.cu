#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeBiasedDistanceKernel(  float *distance, float *biasedDistance, float *biasTerm, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
biasedDistance[threadId] = distance[threadId] + biasTerm[threadId];
}
}