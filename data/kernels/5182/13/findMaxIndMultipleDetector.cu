#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findMaxIndMultipleDetector(float *input, int* maxInd, int size)
{
int maxIndex = 0;
int count = 1;

for (int i = 1; i < size; i++){
if (input[maxIndex] < input[i]){
maxIndex = i;
count = 1;
}
else if (input[maxIndex] == input[i]){
count++;
}
}
if(count>1)
maxInd[0] = -1;
else
maxInd[0] = maxIndex;
}