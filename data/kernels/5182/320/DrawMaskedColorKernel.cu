#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawMaskedColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int targetPixels = targetWidth * targetHeight;

int texturePixels = textureWidth * textureHeight;

int idTextureRgb = id / texturePixels;
int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
int idTextureY = idTexturePixel / textureWidth;
int idTextureX = (idTexturePixel - idTextureY * textureWidth); // same as (id % textureWidth), but the kernel runs another 10% faster

if (idTextureRgb < 3) // only RGB channels are interesting
{
// if the texture pixel offset by inputX, inputY, lies inside the target
if (idTextureX + inputX < targetWidth &&
idTextureX + inputX >= 0 &&
idTextureY + inputY < targetHeight &&
idTextureY + inputY >= 0)
{
int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
float a = textureMask[aIndex];

if (a > 0) // mask allows color here
{
switch (idTextureRgb)
{
case 0:
target[tIndex] = r;
break;
case 1:
target[tIndex] = g;
break;
case 2:
default:
target[tIndex] = b;
break;
}
}
}
}
}