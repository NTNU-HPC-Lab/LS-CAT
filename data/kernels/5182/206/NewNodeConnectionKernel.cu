#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NewNodeConnectionKernel(  int f, int q, int r, int *activityFlag, int *connection, int *age, float *localError, float alfa, int maxCells, float errorFraction  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
activityFlag[r] = 1;

connection[q * maxCells + f] = 0;
age[q * maxCells + f] = 0;
connection[f * maxCells + q] = 0;
age[f * maxCells + q] = 0;
connection[q * maxCells + r] = 1;
age[q * maxCells + r] = 0;
connection[r * maxCells + q] = 1;
age[r * maxCells + q] = 0;
connection[f * maxCells + r] = 1;
age[f * maxCells + r] = 0;
connection[r * maxCells + f] = 1;
age[r * maxCells + f] = 0;

localError[q] -= alfa * localError[q];
localError[f] -= alfa * localError[f];

localError[r] = errorFraction * (localError[q] + localError[f]);
}
}