#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "LSTMDeltaKernelBPTT.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *deltas = NULL;
hipMalloc(&deltas, XSIZE*YSIZE);
float *cellStates = NULL;
hipMalloc(&cellStates, XSIZE*YSIZE);
float *previousCellStates = NULL;
hipMalloc(&previousCellStates, XSIZE*YSIZE);
float *cellStateErrors = NULL;
hipMalloc(&cellStateErrors, XSIZE*YSIZE);
float *nextCellStateErrors = NULL;
hipMalloc(&nextCellStateErrors, XSIZE*YSIZE);
float *outputGateDeltas = NULL;
hipMalloc(&outputGateDeltas, XSIZE*YSIZE);
float *forgetGateDeltas = NULL;
hipMalloc(&forgetGateDeltas, XSIZE*YSIZE);
float *nextForgetGateDeltas = NULL;
hipMalloc(&nextForgetGateDeltas, XSIZE*YSIZE);
float *inputGateDeltas = NULL;
hipMalloc(&inputGateDeltas, XSIZE*YSIZE);
float *nextInputGateDeltas = NULL;
hipMalloc(&nextInputGateDeltas, XSIZE*YSIZE);
float *cellInputDeltas = NULL;
hipMalloc(&cellInputDeltas, XSIZE*YSIZE);
float *cellInputActivations = NULL;
hipMalloc(&cellInputActivations, XSIZE*YSIZE);
float *cellStateActivations = NULL;
hipMalloc(&cellStateActivations, XSIZE*YSIZE);
float *outputGateActivations = NULL;
hipMalloc(&outputGateActivations, XSIZE*YSIZE);
float *nextForgetGateActivations = NULL;
hipMalloc(&nextForgetGateActivations, XSIZE*YSIZE);
float *inputGateActivations = NULL;
hipMalloc(&inputGateActivations, XSIZE*YSIZE);
float *cellInputActivationDerivatives = NULL;
hipMalloc(&cellInputActivationDerivatives, XSIZE*YSIZE);
float *cellStateActivationDerivatives = NULL;
hipMalloc(&cellStateActivationDerivatives, XSIZE*YSIZE);
float *outputGateActivationDerivatives = NULL;
hipMalloc(&outputGateActivationDerivatives, XSIZE*YSIZE);
float *forgetGateActivationDerivatives = NULL;
hipMalloc(&forgetGateActivationDerivatives, XSIZE*YSIZE);
float *inputGateActivationDerivatives = NULL;
hipMalloc(&inputGateActivationDerivatives, XSIZE*YSIZE);
float *cellInputWeights = NULL;
hipMalloc(&cellInputWeights, XSIZE*YSIZE);
float *outputGateWeights = NULL;
hipMalloc(&outputGateWeights, XSIZE*YSIZE);
float *forgetGateWeights = NULL;
hipMalloc(&forgetGateWeights, XSIZE*YSIZE);
float *inputGateWeights = NULL;
hipMalloc(&inputGateWeights, XSIZE*YSIZE);
int inputCount = 1;
int cellCount = 1;
int cellsPerBlock = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
LSTMDeltaKernelBPTT<<<gridBlock,threadBlock>>>(deltas,cellStates,previousCellStates,cellStateErrors,nextCellStateErrors,outputGateDeltas,forgetGateDeltas,nextForgetGateDeltas,inputGateDeltas,nextInputGateDeltas,cellInputDeltas,cellInputActivations,cellStateActivations,outputGateActivations,nextForgetGateActivations,inputGateActivations,cellInputActivationDerivatives,cellStateActivationDerivatives,outputGateActivationDerivatives,forgetGateActivationDerivatives,inputGateActivationDerivatives,cellInputWeights,outputGateWeights,forgetGateWeights,inputGateWeights,inputCount,cellCount,cellsPerBlock);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
LSTMDeltaKernelBPTT<<<gridBlock,threadBlock>>>(deltas,cellStates,previousCellStates,cellStateErrors,nextCellStateErrors,outputGateDeltas,forgetGateDeltas,nextForgetGateDeltas,inputGateDeltas,nextInputGateDeltas,cellInputDeltas,cellInputActivations,cellStateActivations,outputGateActivations,nextForgetGateActivations,inputGateActivations,cellInputActivationDerivatives,cellStateActivationDerivatives,outputGateActivationDerivatives,forgetGateActivationDerivatives,inputGateActivationDerivatives,cellInputWeights,outputGateWeights,forgetGateWeights,inputGateWeights,inputCount,cellCount,cellsPerBlock);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
LSTMDeltaKernelBPTT<<<gridBlock,threadBlock>>>(deltas,cellStates,previousCellStates,cellStateErrors,nextCellStateErrors,outputGateDeltas,forgetGateDeltas,nextForgetGateDeltas,inputGateDeltas,nextInputGateDeltas,cellInputDeltas,cellInputActivations,cellStateActivations,outputGateActivations,nextForgetGateActivations,inputGateActivations,cellInputActivationDerivatives,cellStateActivationDerivatives,outputGateActivationDerivatives,forgetGateActivationDerivatives,inputGateActivationDerivatives,cellInputWeights,outputGateWeights,forgetGateWeights,inputGateWeights,inputCount,cellCount,cellsPerBlock);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}