#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FloorKernel(float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x	+ blockDim.x * blockIdx.x	+ threadIdx.x;
if(id < size)
{
output[id] = floor(input[id]);
}
}