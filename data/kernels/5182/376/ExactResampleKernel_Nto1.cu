#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExactResampleKernel_Nto1(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size = outputWidth * outputHeight;

if (id < size)
{
//output point coordinates
int px = id % outputWidth;
int py = id / outputWidth;

int xRatio = inputWidth / outputWidth;
int yRatio = inputHeight / outputHeight;

float sum = 0;
for (int sx = 0; sx < xRatio; sx++) {
for (int sy = 0; sy < yRatio; sy++) {
//corresponding coordinates in the original image
int x = px * xRatio + sx;
int y = py * yRatio + sy;

sum += input[y * inputWidth + x];
}
}

output[py * outputWidth + px] = sum / (float)(xRatio * yRatio);
}
}