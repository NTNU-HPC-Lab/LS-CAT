#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumNewCentroidCoordinatesKernel( float *input, int imgWidth, int imgHeight, float *centroidCoordinates, int *nearestCentroid, float *pointsWeight, int inputSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
int pointX = threadId % imgWidth;
int pointY = threadId / imgWidth;

float X = (float)pointX;
float Y = (float)pointY;

int centroidId = nearestCentroid[threadId];

float weight = input[threadId];

atomicAdd(&centroidCoordinates[centroidId * 2], weight * X);
atomicAdd(&centroidCoordinates[centroidId * 2 + 1], weight * Y);
atomicAdd(&pointsWeight[centroidId], weight);
}
}