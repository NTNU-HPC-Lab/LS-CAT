#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_MultiplElementWise_naiveOLD(const float * A , int Acount, int Acols, const float * B , int Bcount, int Bcols, float * out0 , int out0count, int out0cols)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int id_row,id_col;
if (id<Acount)
{
if (Acount==Bcount) // matrix .* matrix
{
out0[id] = A[id]*B[id];
}
else if (Bcols==1) // matrix .* row vector
{
id_row = id/Acols;
out0[id] = A[id]*B[id_row];
}
else // matrix .* column vector
{
id_col = id%Acols;
out0[id] = A[id]*B[id_col];
}
}
}