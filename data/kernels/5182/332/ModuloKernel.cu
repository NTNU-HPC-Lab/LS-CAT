#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ModuloKernel(float* input, int divisor, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x	+ blockDim.x * blockIdx.x	+ threadIdx.x;
if(id < size)
{
output[id] = (float)   (((int)input[id]) % divisor) ;
}
}