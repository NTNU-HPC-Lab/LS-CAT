#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExactResampleKernel_1toN(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size = outputWidth * outputHeight;

if (id < size)
{
//output point coordinates
int px = id % outputWidth;
int py = id / outputWidth;

int xRatio = outputWidth / inputWidth;
int yRatio = outputHeight / inputHeight;

//corresponding coordinates in the original image
int x = px / xRatio;
int y = py / yRatio;

output[py * outputWidth + px] = input[y * inputWidth + x];
}
}