#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CompareVectorsKernel(float* inputOne, float* inputTwo, float* output)
{
int id = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (inputOne[id] != inputTwo[id])
output[0] = 1;
}