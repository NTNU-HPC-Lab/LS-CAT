#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizePositionKernel( float *input, float *normalized, float xMax, float yMax )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
normalized[0] = input[0] / xMax;
normalized[1] = input[1] / yMax;
}
}