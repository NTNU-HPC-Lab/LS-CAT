#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetForcesToZeroKernel(  float *force, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells * 3)
{
force[threadId] = 0.00f;
}
}