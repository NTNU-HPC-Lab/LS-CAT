#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void WinnersKernel(  float *winner, float *vertexData, int vertexOffset, float *pointsCoordinates, float cubeSize, int maxCells )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
if(winner[threadId] == 1.00f)
{
float x = pointsCoordinates[threadId * 3];
float y = pointsCoordinates[threadId * 3 + 1];
float z = pointsCoordinates[threadId * 3 + 2];

float side = 1.2f * cubeSize;
float halfSize = 0.50f * side;

// bottom side
vertexData[vertexOffset] = x - halfSize;
vertexData[vertexOffset + 1] = y - halfSize;
vertexData[vertexOffset + 2] = z - halfSize;

vertexData[vertexOffset + 3] = x - halfSize;
vertexData[vertexOffset + 4] = y - halfSize;
vertexData[vertexOffset + 5] = z + halfSize;


vertexData[vertexOffset + 6] = x + halfSize;
vertexData[vertexOffset + 7] = y - halfSize;
vertexData[vertexOffset + 8] = z + halfSize;

vertexData[vertexOffset + 9] = x + halfSize;
vertexData[vertexOffset + 10] = y - halfSize;
vertexData[vertexOffset + 11] = z - halfSize;
}
}
}