#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_cos_naive(const float * A , int Acount, int Acols, float * out0 , int out0count)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<out0count){
out0[id] = cos(A[id]);
}
}