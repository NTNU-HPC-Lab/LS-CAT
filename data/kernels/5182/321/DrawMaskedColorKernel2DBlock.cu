#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawMaskedColorKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
{
int id = blockDim.x * blockDim.y * (blockIdx.y * gridDim.x + blockIdx.x)
+ blockDim.x * threadIdx.y
+ threadIdx.x; // 2D grid of 2D blocks; block dimension x = texture width;
// grid dimension x + block dimension y = texture height

int targetPixels = targetWidth * targetHeight;

int texturePixels = textureWidth * textureHeight;

int idTextureRgb = blockIdx.y;
int idTexturePixel = (id - idTextureRgb * texturePixels);
int idTextureY = blockIdx.x * blockDim.y + threadIdx.y;
int idTextureX = threadIdx.x;


if (idTextureRgb < 3) // only RGB channels are interesting
{
// if the texture pixel offset by inputX, inputY, lies inside the target
if (idTextureX + inputX < targetWidth &&
idTextureX + inputX >= 0 &&
idTextureY + inputY < targetHeight &&
idTextureY + inputY >= 0)
{
int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
float a = textureMask[aIndex];

if (a > 0) // mask allows color here
{
switch (idTextureRgb)
{
case 0:
target[tIndex] = r;
break;
case 1:
target[tIndex] = g;
break;
case 2:
default:
target[tIndex] = b;
break;
}
}
}
}
}