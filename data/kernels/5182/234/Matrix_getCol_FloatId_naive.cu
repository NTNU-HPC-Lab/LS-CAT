#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_getCol_FloatId_naive(const float * A , int Acount, int Acols, float * out0 , int out0count, int out0cols, float col_id)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<Acount/Acols)
{
out0[id] = A[id*Acols + (int)col_id];
}
}