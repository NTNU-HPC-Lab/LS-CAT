#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMGateGradientKernelBPTT( float *input, float *previousOutput, float *cellStates,  float *inputGateDeltas, float *forgetGateDeltas, float *outputGateDeltas,  float* outputGateWeightGradient, float* inputGateWeightGradient, float* forgetGateWeightGradient,  int inputCount, int previousOutputCount, int cellsPerBlock )
{
int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
{
int fromId = weightId % weightsPerGate;
int toId = weightId / weightsPerGate;

//calculate output gate weight gradient
int isFromInputUnit = fromId >= 0 && fromId < inputCount;
int isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
int isPeephole = (fromId >= inputCount + previousOutputCount) && (fromId < inputCount + previousOutputCount + cellsPerBlock);
int isFromBiasUnit = fromId == (inputCount + previousOutputCount + cellsPerBlock);

float inputFromWeight = isFromInputUnit * input[isFromInputUnit * fromId]
+ isFromPreviousOutputUnit * previousOutput[isFromPreviousOutputUnit * (fromId - inputCount)]
+ isPeephole * cellStates[isPeephole * (toId * cellsPerBlock + (fromId - inputCount - previousOutputCount))]
+ isFromBiasUnit * 1;

outputGateWeightGradient[weightId] = outputGateDeltas[toId] * inputFromWeight;
inputGateWeightGradient[weightId] = inputGateDeltas[toId] * inputFromWeight;
forgetGateWeightGradient[weightId] = forgetGateDeltas[toId] * inputFromWeight;
}
}