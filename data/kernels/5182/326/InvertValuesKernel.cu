#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InvertValuesKernel(float *input, float* outputs, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if(id < size)
{
outputs[id] = 1.00f - input[id];
}
}