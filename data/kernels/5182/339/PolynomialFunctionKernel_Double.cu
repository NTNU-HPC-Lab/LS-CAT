#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PolynomialFunctionKernel_Double(float a3, float a2, float a1, float a0, double* input, double* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (id < size)
{
double x = input[id];
output[id] = a3 * x * x * x + a2 * x * x + a1 * x + a0;
}
}