#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearAddSubImageKernel(float *input, float *opImage, float* subImageDefs, int inputWidth, int inputHeight, int opImageWidth, int opImageHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

float subImgCX = subImageDefs[0]; // <-1, 1>
float subImgCY = subImageDefs[1]; // <-1, 1>
float subImgDiameter = subImageDefs[2]; // <0,1>

int maxDiameter = min(inputWidth, inputHeight);
int diameterPix = (int)(subImgDiameter * maxDiameter);
diameterPix = max(1, diameterPix);

int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

int px = id % diameterPix;
int py = id / diameterPix;

if (px + subImgX >= 0 && py + subImgY >= 0 &&
px + subImgX < inputWidth && py + subImgY < inputHeight &&
py < diameterPix )
{
float xRatio = (float)(opImageWidth - 1) / (diameterPix);
float yRatio = (float)(opImageHeight - 1) / (diameterPix);

int x = (int) (xRatio * px);
int y = (int) (yRatio * py);

// X and Y distance difference
float xDist = (xRatio * px) - x;
float yDist = (yRatio * py) - y;

// Points
float topLeft= opImage[y * opImageWidth + x];
float topRight = opImage[y * opImageWidth + x + 1];
float bottomLeft = opImage[(y + 1) * opImageWidth + x];
float bottomRight = opImage[(y + 1) * opImageWidth + x + 1];

float result =
topLeft * (1 - xDist) * (1 - yDist) +
topRight * xDist * (1 - yDist) +
bottomLeft * yDist * (1 - xDist) +
bottomRight * xDist * yDist;


input[(py + subImgY) * inputWidth + px + subImgX] += result;
}
}