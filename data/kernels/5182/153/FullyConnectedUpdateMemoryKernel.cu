#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedUpdateMemoryKernel( float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
if (!dropoutMaskPtr[j])
{
int index = j;
for (i = 0; i < prevLayerSize; i++)
{
// update memory size
weightMemorySizePtr[index] = (1.0f - avgWeightGradPtr[index] * avgWeightGradPtr[index] / avgWeightGradVarPtr[index]) * weightMemorySizePtr[index] + 1.0f;

index += thisLayerSize;
}

// update memory size
biasMemorySizePtr[j] = (1.0f - avgBiasGradPtr[j] * avgBiasGradPtr[j] / avgBiasGradVarPtr[j]) * biasMemorySizePtr[j] + 1.0f;
}
}
}