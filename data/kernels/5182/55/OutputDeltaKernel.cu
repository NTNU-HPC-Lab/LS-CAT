#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OutputDeltaKernel(float *outputDeltas, float *target, float *outputActivations, float *outputActivationDerivatives)
{
int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;
if (unitId < D_OUTPUT_UNITS)
{
outputDeltas[unitId] = (target[unitId] - outputActivations[unitId]) * outputActivationDerivatives[unitId];
}
}