#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void SetValue(float* im, float val, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size)
im[id] = val;
}