#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int2 & subImg, int & diameterPix)
{
diameterPix = (int)( fminf( (float)inputWidth,(float)inputHeight ) * subImageDefs[2] ); // <0,1>

subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f) ;//- diameterPix / 2;
subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);// - diameterPix / 2;

int maxDiameter = min(inputWidth - 1, inputHeight - 1);

diameterPix = max(1, diameterPix);
diameterPix = min(maxDiameter, diameterPix);

if (safeBounds)
{
subImg.x = max(subImg.x, 1);
subImg.y = max(subImg.y, 1);
subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
subImg.y = min(subImg.y, inputHeight - diameterPix - 1);
}
}
__global__ void RetinaTransform_HaveAtLeastOneValueThere (float * subImageDefs, float* input, int inputWidth, int inputHeight, float* output,int outputDataSize, float* retinaMask, int retinaDataSize, int retinaMaskColHint, float* retinaDataInserted)
{
int id_retinaPoint = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int2 subImg;
int diameterPix;
bool  safeBounds = 0;


EstimateParForSubsample( subImageDefs,  safeBounds, inputWidth,  inputHeight,  subImg, diameterPix );

if (id_retinaPoint<outputDataSize)
{
output[id_retinaPoint] = 0; // default value
float x_mask = (retinaMask[id_retinaPoint*retinaMaskColHint]*diameterPix);
float y_mask = (retinaMask[id_retinaPoint*retinaMaskColHint+1]*diameterPix);

int x = subImg.x + x_mask;
int y = subImg.y + y_mask;
if (x<inputWidth && y<inputHeight && x>=0 && y>=0)
{
float val = input[x+y*inputWidth];
output[id_retinaPoint] = val;

atomicAdd(output + id_retinaPoint , val);
atomicAdd(retinaDataInserted + id_retinaPoint , 1);
}
}
}