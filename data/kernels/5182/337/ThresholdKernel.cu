#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ThresholdKernel(float min, float max, int mode, float* input, float* output, int size, int count)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

__shared__ float delta;

if(id < size)
{
if (threadIdx.x == 0)
delta = (max - min)/count;
__syncthreads();

for (int i = 0; i < count; i++)
output[i * size + id] = 0;

int idx;
float fidx = ((input[id] - min) / delta);

switch (mode)
{
case 0: // consider values outside of the interval <min,max>
idx = (int)floor(fmaxf(0, fminf(fidx, count - 1)));
break;
case 1: // strict threshold
if (fidx < 0.0f || fidx >= count)
{
return;
}
idx = (int)fidx;
break;
}
output[idx * size + id] = 1.0f;
}
}