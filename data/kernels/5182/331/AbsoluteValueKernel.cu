#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AbsoluteValueKernel(float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if(id < size)
{
output[id] = fabsf(input[id]);
}
}