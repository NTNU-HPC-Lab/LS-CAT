#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateCC_XY( float *CCXY, int id_CC, float *XY_tofill, int dim_XY ){
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x	+ threadIdx.x;
if(id < dim_XY)
CCXY[id_CC*dim_XY + id] = XY_tofill[id];
}