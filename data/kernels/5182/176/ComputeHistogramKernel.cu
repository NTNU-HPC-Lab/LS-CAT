#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeHistogramKernel(  float *globalMemData, int *globalHist  )
{
//the kernel should be only 1D
int globalThreadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;
int localThreadId = threadIdx.x;
extern __shared__ int partialHist[];

if(localThreadId < D_BINS)
{
//set the partial histogram in shared memory to zero
partialHist[localThreadId] = 0;
}

__syncthreads();

//if the global thread id is within bounds of the data array size
if(globalThreadId < D_MEMORY_BLOCK_SIZE)
{
//copy the global data to local memory
float myLocalDataValue = globalMemData[globalThreadId];
int binIdToWrite = 0 + (D_BINS - 1) * (myLocalDataValue > D_MAX_VALUE);

//if the local value is within limits
if(myLocalDataValue >= D_MIN_VALUE && myLocalDataValue <= D_MAX_VALUE)
{
float biasedValue = myLocalDataValue - D_MIN_VALUE;

binIdToWrite = (int)floor((double)(biasedValue/D_BIN_VALUE_WIDTH)) + 1;
if(myLocalDataValue == D_MAX_VALUE)
{
binIdToWrite = D_BINS - 2;
}

}
//write to local histogram
atomicAdd( &(partialHist[binIdToWrite]), 1);

__syncthreads();

if(localThreadId < D_BINS)
{
//copy values to global histogam
atomicAdd( &(globalHist[localThreadId]), partialHist[localThreadId]);
}
}
}