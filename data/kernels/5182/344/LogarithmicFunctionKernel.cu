#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LogarithmicFunctionKernel(float* input, float* output, int size, const int type)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (id < size)
{
switch (type)
{
case 1:
output[id] = logf(input[id]);
break;
case 2:
output[id] = log2f(input[id]);
break;
case 3:
output[id] = log10f(input[id]);
break;
}
}
}