#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PrepareMeanStdDev(float* input, float* delta, int imageWidth, int imageHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int size = imageWidth * imageHeight;

if (id < size)
{
int px = id % imageWidth;
int py = id / imageWidth;

float2 pixPos = {  2.0f * px / imageWidth - 1,  2.0f * py / imageHeight - 1};

//mean sum
delta[id] = input[id] * pixPos.x;
delta[id + size] = input[id] * pixPos.y;

//variance sum
delta[id + 2 * size] = input[id] * pixPos.x * pixPos.x;
delta[id + 3 * size] = input[id] * pixPos.y * pixPos.y;
}
}