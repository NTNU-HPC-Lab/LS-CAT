#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Crop2DKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int size, int leftMargin, int topMargin, float fillValue)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (id < size)
{
int inputX = id % outputWidth - leftMargin;
int inputY = id / outputWidth - topMargin;

if (inputX >= 0 && inputX < inputWidth && inputY >= 0 && inputY < inputHeight)
output[id] = input[inputX + inputY * inputWidth];
else
output[id] = fillValue;
}
}