#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeRecurrentWeightsKernel( float *recurrentWeights, float *recurrentWeightDeltas, float *outputWeights, float *outputDeltas, float *recurrentWeightRTRLDerivatives,  float trainingRate, float momentum )
{
int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS)
{
float gradient = 0;

for (int i = 0; i < D_OUTPUT_UNITS; i++)
{
float sum = 0;
for (int j = 0; j < D_HIDDEN_UNITS; j++)
{
sum += outputWeights[i * D_HIDDEN_UNITS + j] * recurrentWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
}

gradient += outputDeltas[i] * sum;
}

float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
recurrentWeightDeltas[weightId] = weightDelta;
recurrentWeights[weightId] += weightDelta;
}
}