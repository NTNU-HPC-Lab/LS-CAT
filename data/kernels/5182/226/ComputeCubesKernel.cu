#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeCubesKernel(  float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, int *activityFlag, int textureWidth, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
float x = pointsCoordinates[threadId * 3];
float y = pointsCoordinates[threadId * 3 + 1];
float z = pointsCoordinates[threadId * 3 + 2];

float halfSide = 0.50f * cubeSide;
if(activityFlag[threadId] == 0)
{
halfSide = 0.00f;
}

int textureOffset = quadOffset + maxCells * 4 * 6 * 3;
float textureAbsLength = (float)(maxCells * textureWidth);


// BOTTOM SIDE
vertexData[quadOffset + threadId * 72] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 1] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 2] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 3] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 4] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 5] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 6] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 7] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 8] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 9] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 10] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 11] = z + halfSide;

vertexData[textureOffset + threadId * 48] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 1] = 0.00f;

vertexData[textureOffset + threadId * 48 + 2] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 3] = 1.00f;

vertexData[textureOffset + threadId * 48 + 4] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 5] = 1.00f;

vertexData[textureOffset + threadId * 48 + 6] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 7] = 0.00f;


// FRONT SIDE
vertexData[quadOffset + threadId * 72 + 12] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 13] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 14] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 15] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 16] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 17] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 18] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 19] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 20] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 21] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 22] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 23] = z + halfSide;



vertexData[textureOffset + threadId * 48 + 8] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 9] = 0.00f;

vertexData[textureOffset + threadId * 48 + 10] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 11] = 1.00f;

vertexData[textureOffset + threadId * 48 + 12] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 13] = 1.00f;

vertexData[textureOffset + threadId * 48 + 14] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 15] = 0.00f;



// LEFT SIDE
vertexData[quadOffset + threadId * 72 + 24] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 25] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 26] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 27] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 28] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 29] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 30] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 31] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 32] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 33] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 34] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 35] = z + halfSide;


vertexData[textureOffset + threadId * 48 + 16] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 17] = 0.00f;

vertexData[textureOffset + threadId * 48 + 18] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 19] = 1.00f;

vertexData[textureOffset + threadId * 48 + 20] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 21] = 1.00f;

vertexData[textureOffset + threadId * 48 + 22] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 23] = 0.00f;

// BACK SIDE
vertexData[quadOffset + threadId * 72 + 36] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 37] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 38] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 39] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 40] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 41] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 42] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 43] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 44] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 45] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 46] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 47] = z - halfSide;


vertexData[textureOffset + threadId * 48 + 24] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 25] = 0.00f;

vertexData[textureOffset + threadId * 48 + 26] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 27] = 1.00f;

vertexData[textureOffset + threadId * 48 + 28] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 29] = 1.00f;

vertexData[textureOffset + threadId * 48 + 30] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 31] = 0.00f;


// RIGHT SIDE
vertexData[quadOffset + threadId * 72 + 48] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 49] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 50] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 51] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 52] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 53] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 54] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 55] = y - halfSide;
vertexData[quadOffset + threadId * 72 + 56] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 57] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 58] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 59] = z + halfSide;

vertexData[textureOffset + threadId * 48 + 32] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 33] = 0.00f;

vertexData[textureOffset + threadId * 48 + 34] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 35] = 1.00f;

vertexData[textureOffset + threadId * 48 + 36] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 37] = 1.00f;

vertexData[textureOffset + threadId * 48 + 38] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 39] = 0.00f;


// UPPER SIDE
vertexData[quadOffset + threadId * 72 + 60] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 61] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 62] = z + halfSide;

vertexData[quadOffset + threadId * 72 + 63] = x - halfSide;
vertexData[quadOffset + threadId * 72 + 64] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 65] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 66] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 67] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 68] = z - halfSide;

vertexData[quadOffset + threadId * 72 + 69] = x + halfSide;
vertexData[quadOffset + threadId * 72 + 70] = y + halfSide;
vertexData[quadOffset + threadId * 72 + 71] = z + halfSide;



vertexData[textureOffset + threadId * 48 + 40] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 41] = 1.00f;

vertexData[textureOffset + threadId * 48 + 42] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 43] = 0.00f;

vertexData[textureOffset + threadId * 48 + 44] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 45] = 0.00f;

vertexData[textureOffset + threadId * 48 + 46] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 48 + 47] = 1.00f;
}



}