#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumSymbolsKernel( float *symbolOne, float *symbolTwo, float *result, int symbolSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < symbolSize)
{
result[threadId] = symbolOne[threadId] + symbolTwo[threadId];
}
}