#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extractCoeffs(const float  *matrices, float *coeffArray, int savedCoeffs, int dimsize){
int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
+ threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

int offsetMatrix = threadGlobalID * dimsize * dimsize,
offsetCoeff = threadGlobalID * savedCoeffs,
coeffsLeft = savedCoeffs,
x, y, y_n = 0, x_n = 1,
numberinrow, tmp;

coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix];
coeffsLeft -= 1;

while (coeffsLeft > 0){
// Work out number in row
x = x_n;
y = y_n;

if (x_n < dimsize - 1)
numberinrow = x_n + 1;
else
numberinrow = x_n - (y_n - 1);

if (numberinrow % 2 == 0){
// Even
while (numberinrow > 0 && coeffsLeft > 0){
coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
numberinrow--;
coeffsLeft--;

if ((numberinrow + 1) % 2 == 0){
// Swap x and y
tmp = x;
x = y;
y = tmp;
}
else{
// Swap x and y
tmp = x;
x = y;
y = tmp;
x--;
y++;
}
}
}
else{
// Odd
while (numberinrow > 1 && coeffsLeft > 0){
coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
numberinrow--;
coeffsLeft--;
if ((numberinrow + 1) % 2 == 1){
// Swap x and y
tmp = x;
x = y;
y = tmp;
}
else{
// Swap x and y
tmp = x;
x = y;
y = tmp;
x--;
y++;
}
}
if (coeffsLeft > 0){
// add the odd one
coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
numberinrow--;
coeffsLeft--;
}
}
if (x_n == dimsize - 1){
y_n++;
}
else{
x_n++;
}
}
}