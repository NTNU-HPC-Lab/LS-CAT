#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CutSubImageKernel_SingleParams(float *input, float *output, float subImageX, float subImageY, float subImageDiameter, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size = outputWidth * outputHeight;

if (id < size)
{
float subImgCX = subImageX; // <-1, 1>
float subImgCY = subImageY; // <-1, 1>
float subImgDiameter = subImageDiameter; // <0,1>

int maxDiameter = min(inputWidth - 1, inputHeight - 1);
int diameterPix = (int)(subImgDiameter * maxDiameter);

diameterPix = max(1, diameterPix);
diameterPix = min(maxDiameter, diameterPix);

int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

if (safeBounds)
{
subImgX = max(subImgX, 1);
subImgY = max(subImgY, 1);

subImgX = min(subImgX, inputWidth - diameterPix - 1);
subImgY = min(subImgY, inputHeight - diameterPix - 1);
}

int px = id % outputWidth;
int py = id / outputWidth;
//
float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);
//
int x = (int)(xRatio * px);
int y = (int)(yRatio * py);

if (x + subImgX >= 0 && y + subImgY >= 0 &&
x + subImgX < inputWidth && y + subImgY < inputHeight)
{
output[py * outputWidth + px] = input[(y + subImgY) * inputWidth + x + subImgX];
}
}
}