#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedShiftKernel( float *weightPtr, float *biasPtr, float *shiftedWeightsPtr, float *shiftedBiasPtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
if (!dropoutMaskPtr[j])
{
// weight gradient
int index = j;
for (i = 0; i < prevLayerSize; i++)
{
shiftedWeightsPtr[index] = weightPtr[index] + avgWeightGradPtr[index]; // TODO: Check if it is correct to add here, or if it should be subtracted
index += thisLayerSize;
}

// bias gradient
shiftedBiasPtr[j] = biasPtr[j] - avgBiasGradPtr[j];
}
}
}