#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetVauleInIdxMinMax( float* vector, int id_min, int id_max, float value)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id >= id_min && id <= id_max)
vector[id] = value;
}