#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RBMInputForwardKernel( float *inputPtr, float *outputPtr, float *biasPtr, bool applyBias, int thisLayerSize )
{
// i: current neuron id
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < thisLayerSize)
{
float result = inputPtr[i];
if (applyBias)
result += biasPtr[i];
outputPtr[i] = inputPtr[i];
}
}