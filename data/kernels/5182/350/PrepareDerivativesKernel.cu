#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PrepareDerivativesKernel(float* input, float* lastInput, float* derivatives, int inputWidth, int inputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size =  inputWidth * inputHeight;

if (id < size)
{
float mul = 100000;
//I_x, I_y
float I_x = mul * derivatives[id];
float I_y = mul * derivatives[size + id];

//I_t
float input_dt = mul * (input[id] - lastInput[id]);
lastInput[id] = input[id];

// I_x * I_y
derivatives[2 * size + id] = I_x * I_y;
// I_x * I_t
derivatives[3 * size + id] = I_x * input_dt;
// I_x * I_t
derivatives[4 * size + id] = I_y * input_dt;
// I_x ^ 2
derivatives[id] = I_x * I_x;
// I_y ^ 2
derivatives[size + id] = I_y * I_y;
}
}