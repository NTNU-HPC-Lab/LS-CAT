#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ElementwiseNorm(float * A, float  *B, int size) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size)
A[id] /= B[id];
}