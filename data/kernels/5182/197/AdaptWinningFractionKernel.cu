#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AdaptWinningFractionKernel(  int s1, float *winningFraction, int *winningCount, float bParam, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
winningFraction[threadId] = winningFraction[threadId] + bParam * ((float)(threadId == s1) - winningFraction[threadId]);
winningCount[threadId] = winningCount[threadId] + (threadId == s1) * 1;
}
}