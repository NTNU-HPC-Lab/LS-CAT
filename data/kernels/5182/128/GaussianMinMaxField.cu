#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianMinMaxField(float* input, int inputCount, float* mins, float* maxes)
{
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < inputCount)
{
mins[i] = fminf(mins[i], input[i]);
maxes[i] = fmaxf(maxes[i], input[i]);
}
}