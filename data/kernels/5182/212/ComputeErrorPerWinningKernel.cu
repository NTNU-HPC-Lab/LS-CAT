#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeErrorPerWinningKernel(  float *localError, int *winningCount, float *errorPerWinning, int *activityFlag, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;


// TO DO: GET RID OF IF-ELSE
if(threadId < maxCells)
{
if(activityFlag[threadId] == 1)
{
if(winningCount[threadId] != 0)
{
errorPerWinning[threadId] = localError[threadId] / (float)winningCount[threadId];
}
else
{
errorPerWinning[threadId] = 0.00f;
}
}
}
}