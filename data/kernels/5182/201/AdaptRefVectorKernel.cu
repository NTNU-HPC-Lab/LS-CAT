#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AdaptRefVectorKernel(  int cell, float *referenceVector, float oldErrorFraction, float youngErrorFraction, float decayFactor, int *winningCount, float *difference, int inputSize  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
float errorFraction = (youngErrorFraction - oldErrorFraction) * expf( - decayFactor * winningCount[cell] ) + oldErrorFraction;
referenceVector[cell * inputSize + threadId] += errorFraction * difference[cell * inputSize + threadId];
}
}