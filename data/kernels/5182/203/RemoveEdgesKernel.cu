#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RemoveEdgesKernel(  int *connection, int *age, int maxAge, int *activityFlag, float *winningFraction, int *winningCount, float *utility, float *localError, int *neuronAge, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
if(activityFlag[threadId] == 1)
{
neuronAge[threadId] = neuronAge[threadId] + 1;

// TO DO : GET RID OF  IFs & ELSEs
int activeConnections = 0;
int connId;
for(int c = 0; c < maxCells; c++)
{
connId = threadId * maxCells + c;
if(connection[connId] == 1)
{
if(age[connId] <= maxAge)
{
activeConnections++;
}
else
{
connection[connId] = 0;
age[connId] = 0;
}
}
}
if(activeConnections == 0)
{
activityFlag[threadId] = 0;
localError[threadId] = 0.00f;
neuronAge[threadId] = 0;
winningFraction[threadId] = 0.00f;
winningCount[threadId] = 0;
utility[threadId] = 0.00f;
}
}
}
}