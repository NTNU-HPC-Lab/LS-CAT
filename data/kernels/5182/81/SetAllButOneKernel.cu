#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetAllButOneKernel(float *buffer, float value, int index, int count)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < count && threadId != index)
{
buffer[threadId] = value;
}

}