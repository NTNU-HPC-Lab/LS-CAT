#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExpKernel_naive(const float * A , int Acount, int Acols, float * out0 , int out0count)
{
int id = blockDim.x * blockIdx.y * gridDim.x	+ blockDim.x * blockIdx.x	+ threadIdx.x;
if(id < out0count)
{
out0[id] = exp(A[id]);
}
}