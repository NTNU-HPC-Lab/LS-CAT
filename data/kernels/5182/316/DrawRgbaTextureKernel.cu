#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaTextureKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int targetPixels = targetWidth * targetHeight;

int texturePixels = textureWidth * textureHeight;

int idTextureRgb = id / texturePixels;
int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
int idTextureY = idTexturePixel / textureWidth;
int idTextureX = (idTexturePixel - idTextureY * textureWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


if (idTextureRgb < 3) // 3 channels that we will write to
{
// the texture is in BGR format, we want RGB
switch (idTextureRgb)
{
case 0: // R
idTextureRgb = 2; // B
break;
case 2: // B
idTextureRgb = 0; // R
break;
}
// if the texture pixel offset by inputX, inputY, lies inside the target
if (idTextureX + inputX < targetWidth &&
idTextureX + inputX >= 0 &&
idTextureY + inputY < targetHeight &&
idTextureY + inputY >= 0)
{
int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
float a = texture[aIndex];
target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
}
}
}