#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaTextureKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int targetPixels = targetWidth * targetHeight;

int texturePixels = textureWidth * textureHeight;

int objectPixels = objectWidth * objectHeight;

int idObjectRgb = id / objectPixels;
int idObjectPixel = (id - idObjectRgb * objectPixels); // same as (id % objectPixels), but the kernel runs 10% faster
int idObjectY = idObjectPixel / objectWidth;
int idObjectX = (idObjectPixel - idObjectY * objectWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


if (idObjectRgb < 3) // 3 channels that we will write to
{
int targetRgb = idObjectRgb;
// the texture is in BGR format, we want RGB
switch (idObjectRgb)
{
case 0: // R
targetRgb = 2; // B
break;
case 2: // B
targetRgb = 0; // R
break;
}
// if the object pixel offset by inputX, inputY, lies inside the target
if (idObjectX + inputX < targetWidth &&
idObjectX + inputX >= 0 &&
idObjectY + inputY < targetHeight &&
idObjectY + inputY >= 0)
{
// nearest neighbor texture X,Y:
int textureX = textureWidth * idObjectX / objectWidth;
int textureY = textureHeight * idObjectY / objectHeight;
int textureId = textureY * textureWidth + textureX;

int rgbIndex = textureId + idObjectRgb * texturePixels;
float textureValue = texture[rgbIndex];

int tIndex = targetPixels * targetRgb + targetWidth * (idObjectY + inputY) + (idObjectX + inputX);
int aIndex = textureId + 3 * texturePixels; // the A component of the texture
float a = texture[aIndex];
target[tIndex] = target[tIndex] * (1.0f - a) + a * textureValue;
}
}
}