#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_sin_naive(const float * A , int Acount, int Acols, float * out0 , int out0count)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<out0count){
out0[id] = sin(A[id]);
}
}