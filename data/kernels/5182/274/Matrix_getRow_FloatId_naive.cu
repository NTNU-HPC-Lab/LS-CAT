#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_getRow_FloatId_naive(const float * A, int Acount, int Acols, float * out0, int out0count, int out0cols, float row_id) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < Acols) {
out0[id] = A[id + (int)row_id*Acols];
}
}