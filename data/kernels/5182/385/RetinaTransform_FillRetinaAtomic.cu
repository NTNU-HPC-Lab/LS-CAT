#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int2 & subImg, int & diameterPix)
{
diameterPix = (int)( fminf( (float)inputWidth,(float)inputHeight ) * subImageDefs[2] ); // <0,1>

subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f) ;//- diameterPix / 2;
subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);// - diameterPix / 2;

int maxDiameter = min(inputWidth - 1, inputHeight - 1);

diameterPix = max(1, diameterPix);
diameterPix = min(maxDiameter, diameterPix);

if (safeBounds)
{
subImg.x = max(subImg.x, 1);
subImg.y = max(subImg.y, 1);
subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
subImg.y = min(subImg.y, inputHeight - diameterPix - 1);
}
}
__global__ void RetinaTransform_FillRetinaAtomic (float * subImageDefs, float* input, int inputWidth, int inputHeight, float* output,int outputDataSize, float* retinaMask, int retinaDataSize, int retinaMaskColHint, float* retinaDataInserted)
{
int id_pxl = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int2 subImg;
int diameterPix;
bool  safeBounds = 0;

int x = id_pxl % inputWidth;
int y = id_pxl/inputWidth;

EstimateParForSubsample( subImageDefs,  safeBounds, inputWidth,  inputHeight,  subImg, diameterPix );

if (id_pxl<inputWidth*inputHeight)
{
float minDist = 999999.9; // ??>? should be written bette
int minIdx = 1;
for (int id_retinaPoint=0 ; id_retinaPoint<retinaDataSize ; id_retinaPoint++)
{
float x_mask = (retinaMask[id_retinaPoint*retinaMaskColHint]*diameterPix);
float y_mask = (retinaMask[id_retinaPoint*retinaMaskColHint+1]*diameterPix);

x_mask += subImg.x;
y_mask += subImg.y;

float dist = (x-x_mask)*(x-x_mask) + (y-y_mask)*(y-y_mask);

if (dist<minDist)
{
minDist = dist;
minIdx  = id_retinaPoint;
}
}
atomicAdd(output + minIdx , input[id_pxl]);
atomicAdd(retinaDataInserted + minIdx , 1);
}
}