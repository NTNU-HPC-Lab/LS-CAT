#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Prepare_1_MeansForJoin(float* input, int c_src1, int c_src2, int c_n, float* delta, int imageWidth, int imageHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int size = imageWidth * imageHeight;

if (id < size)
{
int px = id % imageWidth;
int py = id / imageWidth;

bool insideSrc1 = delta[c_src1 * NUM_SUMS * size + 4 * size + id] != 0;
bool insideSrc2 = delta[c_src2 * NUM_SUMS * size + 4 * size + id] != 0;

if (input[id] > 0 && (insideSrc1 || insideSrc2)) {

float2 pixPos = {  2.0f * px / imageWidth - 1,  2.0f * py / imageHeight - 1};

//w * pos
delta[c_n * NUM_SUMS * size + 0 * size + id] = input[id] * pixPos.x;
delta[c_n * NUM_SUMS * size + 1 * size + id] = input[id] * pixPos.y;

//w * pos^2
delta[c_n * NUM_SUMS * size + 2 * size + id] = input[id] * pixPos.x * pixPos.x;
delta[c_n * NUM_SUMS * size + 3 * size + id] = input[id] * pixPos.y * pixPos.y;

//w
delta[c_n * NUM_SUMS * size + 4 * size + id] = input[id];
}
else
{
delta[c_n * NUM_SUMS * size + 0 * size + id] = 0;
delta[c_n * NUM_SUMS * size + 1 * size + id] = 0;

//w * pos^2
delta[c_n * NUM_SUMS * size + 2 * size + id] = 0;
delta[c_n * NUM_SUMS * size + 3 * size + id] = 0;

//w
delta[c_n * NUM_SUMS * size + 4 * size + id] = 0;
}
}
}