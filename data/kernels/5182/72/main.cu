#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Prepare_1_MeansForJoin.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *input = NULL;
hipMalloc(&input, XSIZE*YSIZE);
int c_src1 = 1;
int c_src2 = 1;
int c_n = 1;
float *delta = NULL;
hipMalloc(&delta, XSIZE*YSIZE);
int imageWidth = XSIZE;
int imageHeight = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Prepare_1_MeansForJoin<<<gridBlock,threadBlock>>>(input,c_src1,c_src2,c_n,delta,imageWidth,imageHeight);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Prepare_1_MeansForJoin<<<gridBlock,threadBlock>>>(input,c_src1,c_src2,c_n,delta,imageWidth,imageHeight);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Prepare_1_MeansForJoin<<<gridBlock,threadBlock>>>(input,c_src1,c_src2,c_n,delta,imageWidth,imageHeight);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}