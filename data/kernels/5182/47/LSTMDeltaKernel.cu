#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMDeltaKernel( float *cellStateErrors, float *outputGateDeltas, float *cellStates, float *outputGateActivations, float *outputGateActivationDerivatives, float *deltas,  int cellCount, int cellsPerBlock )
{
int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (memoryBlockId < cellCount / cellsPerBlock)
{
float outputGateDeltaSum = 0.0;

for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
{
float delta = deltas[cellId];
cellStateErrors[cellId] = outputGateActivations[memoryBlockId] * delta;
outputGateDeltaSum += cellStates[cellId] * delta;
}

outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;
}
}