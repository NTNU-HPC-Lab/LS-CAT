#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RecurrentWeightsRTRLDerivativesKernel( float *previousHiddenActivations, float *hiddenActivationDerivatives, float *recurrentWeights, float *recurrentWeightRTRLDerivatives, float *previousRecurrentWeightRTRLDerivatives )
{
int partialId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS)
{
int unitId = partialId / (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
int weightId = partialId % (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
int to = weightId / D_HIDDEN_UNITS;
int from = weightId % D_HIDDEN_UNITS;

float sum = 0;
for (int i = 0; i < D_HIDDEN_UNITS; i++)
{
sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousRecurrentWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_HIDDEN_UNITS) + weightId];
}

recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * previousHiddenActivations[from] + sum);
}
}