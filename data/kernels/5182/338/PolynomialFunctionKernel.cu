#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PolynomialFunctionKernel(float a3, float a2, float a1, float a0, float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if(id < size)
{
float x = input[id];
output[id] = a3 * x * x * x + a2 * x * x + a1 * x + a0;
}
}