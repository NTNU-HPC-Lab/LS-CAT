#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleSubImageKernel(float *input, float *output, float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size =  outputWidth * outputHeight;

if (id < size)
{
float subImgCX = subImageDefs[0]; // <-1, 1>
float subImgCY = subImageDefs[1]; // <-1, 1>
float subImgDiameter = subImageDefs[2]; // <0,1>

int maxDiameter = min(inputWidth - 1, inputHeight - 1);
int diameterPix = (int)(subImgDiameter * maxDiameter);

diameterPix = max(1, diameterPix);
diameterPix = min(maxDiameter, diameterPix);

int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

if (safeBounds)
{
subImgX = max(subImgX, 1);
subImgY = max(subImgY, 1);

subImgX = min(subImgX, inputWidth - diameterPix - 1);
subImgY = min(subImgY, inputHeight - diameterPix - 1);
}

int px = id % outputWidth;
int py = id / outputWidth;

float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

int x = (int) (xRatio * px);
int y = (int) (yRatio * py);

if (x + subImgX >= 0 && y + subImgY >= 0 &&
x + subImgX < inputWidth && y + subImgY < inputHeight)
{
// X and Y distance difference
float xDist = (xRatio * px) - x;
float yDist = (yRatio * py) - y;

// Points
float topLeft= input[(y + subImgY) * inputWidth + x + subImgX];
float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1];

float result =
topLeft * (1 - xDist) * (1 - yDist) +
topRight * xDist * (1 - yDist) +
bottomLeft * yDist * (1 - xDist) +
bottomRight * xDist * yDist;

output[py * outputWidth + px] = result;
}
}
}