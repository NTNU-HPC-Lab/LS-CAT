#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LengthFromElements(float* element1, float* element2, float* output, int count)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < count)
{
output[threadId] = sqrtf(element1[threadId] * element1[threadId] + element2[threadId] * element2[threadId]);
}
}