#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetKernel(float *buffer, int offset, float value, int count)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < count)
{
(buffer + offset)[threadId] = value;
}

}