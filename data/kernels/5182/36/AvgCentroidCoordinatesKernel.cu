#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AvgCentroidCoordinatesKernel(  float *centroidCoordinates, float *pointsWeight, int inputSize, int centroids )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < centroids * 2)
{
if(pointsWeight[threadId / 2] == 0.00f)
{
centroidCoordinates[threadId] = 0.00f;
}
else
{
centroidCoordinates[threadId] = centroidCoordinates[threadId] / pointsWeight[threadId / 2];
}
}
}