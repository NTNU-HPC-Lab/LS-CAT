#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitCentroidsKernel(  float *centroidCoordinates, float *randomNumbers, float minX, float maxX, float minY, float maxY, int centroids  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < centroids)
{
centroidCoordinates[threadId *2] = (maxX - minX) * randomNumbers[threadId * 2] + minX;
centroidCoordinates[threadId * 2 + 1] = (maxY - minY) * randomNumbers[threadId * 2 + 1] + minY;
}
}