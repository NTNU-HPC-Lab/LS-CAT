#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddUtilityKernel(  int s1, int s2, float *distance, float *utility  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
utility[s1] += distance[s2] - distance[s1];
}
}