#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyBrainsMovement( float *CCXY, int dim_XY, float *movement, int dim_movement, int max_clusters ){
int id = blockDim.x*blockIdx.y*gridDim.x   + blockDim.x*blockIdx.x   + threadIdx.x;
if (id<max_clusters){
//--- move in XY
if (dim_movement>=2){
CCXY[id*dim_XY]   -= movement[0];
CCXY[id*dim_XY+1] -= movement[1];
}
//--- apply rotation in X
if (dim_movement>=3){
}
}
}