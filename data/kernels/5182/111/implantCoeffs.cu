#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void implantCoeffs(float* matrices, float *coeffArray, int savedCoeffs, int dimsize){

int id = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
+ threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

int offsetMatrix = id * dimsize * dimsize,
offsetCoeff = id * savedCoeffs,
coeffsLeft = savedCoeffs,
x, y, y_n = 0, x_n = 1,
numberinrow, tmp;

matrices[offsetMatrix] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
coeffsLeft -= 1;

while (coeffsLeft > 0){
// Work out number in row
x = x_n;
y = y_n;

if (x_n < dimsize - 1){
numberinrow = x_n + 1;
}
else{
numberinrow = x_n - (y_n - 1);
}

if (numberinrow % 2 == 0){
// Even
while (numberinrow > 0 && coeffsLeft > 0){
matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
numberinrow--;
coeffsLeft--;

if ((numberinrow + 1) % 2 == 0){
// Swap x and y
tmp = x;
x = y;
y = tmp;
}
else{
// Swap x and y
tmp = x;
x = y;
y = tmp;
x--;
y++;
}
}
}
else{
// Odd
while (numberinrow > 1 && coeffsLeft > 0){
matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
numberinrow--;
coeffsLeft--;
if ((numberinrow + 1) % 2 == 1){
// Swap x and y
tmp = x;
x = y;
y = tmp;
}
else{
// Swap x and y
tmp = x;
x = y;
y = tmp;
x--;
y++;
}
}
if (coeffsLeft > 0){
// add the odd one
matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
numberinrow--;
coeffsLeft--;
}
}
if (x_n == dimsize - 1){
y_n++;
}
else{
x_n++;
}
}

}