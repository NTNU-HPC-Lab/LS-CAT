#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CenterOfGravityKernel(  float *pointsCoordinates, float *centerOfGravity, int *activityFlag, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
float xSum = 0.00f, ySum = 0.00f, zSum = 0.00f;
int livingCells = 0;
for(int c = 0; c < maxCells; c++)
{
if(activityFlag[c] == 1)
{
xSum += pointsCoordinates[c * 3];
ySum += pointsCoordinates[c * 3 + 1];
zSum += pointsCoordinates[c * 3 + 2];

livingCells++;
}
}
centerOfGravity[0] = xSum / (float)livingCells;
centerOfGravity[1] = ySum / (float)livingCells;
centerOfGravity[2] = zSum / (float)livingCells;
}
}