#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NegativeCorrelationForwardResetKernel( float* outputPtr, int thisLayerSize )
{
// j: current layer neuron id
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
outputPtr[j] = 0;
}
}