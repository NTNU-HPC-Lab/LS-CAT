#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NegativeCorrelationForwardSumKernel( float* inputPtr, float* outputPtr, int thisLayerSize )
{
// j: current layer neuron id
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
outputPtr[j] += inputPtr[j];
}
}