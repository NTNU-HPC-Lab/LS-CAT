#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void OFConvertXY2AngleSize (float*of, int id, int imageSize, float& of_size, float& of_angle){
float2 OF_value;

OF_value.x = of[id];
OF_value.y = of[id+imageSize];

of_size  = (float) sqrt( (OF_value.x+OF_value.y) * (OF_value.x+OF_value.y) );  // normalized to be <0,1>
of_angle = (float) atan2(OF_value.x,OF_value.y);  // <-PI;PI>
}
__global__ void OFConvert2AngleSize (float*of, int imageSize){
int id = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

float OF_size;
float OF_angle;

if (id<imageSize){
OFConvertXY2AngleSize(of,id,imageSize,OF_size,OF_angle);

of[id] = OF_angle;
of[id+imageSize] = OF_size;
}
}