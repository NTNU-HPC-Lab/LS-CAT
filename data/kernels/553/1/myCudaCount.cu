#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myCudaCount(const char *text, int *pos, int n){
int bid = blockIdx.x;
int left = (blockIdx.y == 1)? bid * blockDim.x + ThreadSize/2: bid * blockDim.x;
int tid = threadIdx.x;
int id = left + tid;

__shared__ int BIT[ThreadSize][LOGT];
__shared__ int iBIT[ThreadSize];
int seqIndex = id * SeqSize;
int seq[SeqSize];

if (seqIndex < n){
// Transform
int count = 1;
for (int i=0; i<SeqSize; i++, count++){
if (seqIndex + i < n){
if (text[seqIndex + i] <= ' ')
count = 0;
}
else{
count = 0;
}
seq[i] = count;
}

iBIT[tid] = SeqSize - 1 - seq[SeqSize - 1];
BIT[tid][0] = seq[SeqSize - 1] / SeqSize;
__syncthreads();

// Build tree
int before = BIT[tid][0];
for (int i=1, offset=1; i<LOGT; i++, offset <<= 1){
int tmp = tid - offset;
if (tmp >= 0){
if (before != 0 && BIT[tmp][i-1] != 0){
before = (BIT[tid][i] = before + BIT[tmp][i-1]);
}
else
before = (BIT[tid][i] = 0);
}
else{
BIT[tid][i] = before;
}
__syncthreads();
}

// Set
int offset = tid - 1;
for (int i=LOGT-1; i>=0 && offset>=0; i--)
offset -= BIT[offset][i];

if (offset >= 0) offset = (left + offset) * SeqSize + iBIT[offset];

if (tid >= ThreadSize/2 || id < ThreadSize / 2){
for (int i=0; i<SeqSize && seqIndex+i<n; i++){
if (seq[i] == 0)
offset = seqIndex + i;
pos[seqIndex + i] = seqIndex + i - offset;
}
}
}
}