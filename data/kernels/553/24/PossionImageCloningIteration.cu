#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PossionImageCloningIteration( const float *fixed, const float *mask, float *input, float *output, const int wt, const int ht ){
const int dir[4][2] = {	{0, -1}, {1, 0}, {0, 1}, {-1, 0}};
int num = 4;
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt * yt + xt;
if (yt < ht && xt < wt && mask[curt] > 127.0f){
float sum[3] = {0};
for (int i=0; i<4; i++){
int dxt = xt + dir[i][0];
int dyt = yt + dir[i][1];
int dcurt = wt * dyt + dxt;
if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht &&
mask[dcurt] > 127.0f){
sum[0] += input[dcurt*3+0];
sum[1] += input[dcurt*3+1];
sum[2] += input[dcurt*3+2];
}
}
output[curt*3+0] = fixed[curt*3+0] + sum[0] / num;
output[curt*3+1] = fixed[curt*3+1] + sum[1] / num;
output[curt*3+2] = fixed[curt*3+2] + sum[2] / num;
}
}