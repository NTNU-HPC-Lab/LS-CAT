#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateSampleT( const float *target, const float *mask, float *subT, int *subM, const int wt, const int ht, const int ws, const int hs, const int sRate ){
const int ys = blockIdx.y * blockDim.y + threadIdx.y;
const int xs = blockIdx.x * blockDim.x + threadIdx.x;
const int curst = ws * ys + xs;

if (ys < hs && xs < ws){
const int yt = ys * sRate;
const int xt = xs * sRate;
int num = 0;
float sum[3] = {0};
int _or = 0;

for (int i=0; i<sRate; i++){
for (int j=0; j<sRate; j++){
if (yt + i < ht && xt + j < wt){
int curt = wt * (yt+i) + (xt+j);
sum[0] += target[curt*3+0];
sum[1] += target[curt*3+1];
sum[2] += target[curt*3+2];
_or |= (mask[curt] > 127.0f);
num++;
}
}
}

subM[curst] = _or;
subT[curst*3+0] += sum[0] / num;
subT[curst*3+1] += sum[1] / num;
subT[curst*3+2] += sum[2] / num;
}
}