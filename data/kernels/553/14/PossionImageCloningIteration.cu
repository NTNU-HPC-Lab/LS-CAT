#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PossionImageCloningIteration( const float *fixed, const float *mask, float *input, float *output, const int wt, const int ht, const int round ){
const int dir[16][2] = {{-2, -2},				{0, -2},			{2, -2},
{-1, -1},	{0, -1},	{1, -1},
{-2, 0},	{-1, 0},				{1, 0},	{2, 0},
{-1, 1},	{0, 1},		{1, 1},
{-2, 2},				{0, 2},				{2, 2}};
const int coef[16] = {	1,		1,		1,
2,	2,	2,
1,	2,		2,	1,
2,	2,	2,
1,		1,		1};
const int num = 24;
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt * yt + xt;
if (yt < ht && xt < wt && mask[curt] > 127.0f){
float sum[3] = {0};
for (int i=0; i<16; i++){
int dxt = xt + dir[i][0];
int dyt = yt + dir[i][1];
int dcurt = wt * dyt + dxt;
if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht &&
mask[dcurt] > 127.0f){
sum[0] += input[dcurt*3+0] * coef[i];
sum[1] += input[dcurt*3+1] * coef[i];
sum[2] += input[dcurt*3+2] * coef[i];
}
}
output[curt*3+0] = fixed[curt*3+0] + sum[0] / num;
output[curt*3+1] = fixed[curt*3+1] + sum[1] / num;
output[curt*3+2] = fixed[curt*3+2] + sum[2] / num;
}
}