#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateFixed( const float *background, const float *target, const float *mask, float *fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox ){
const int dir[16][2] = {{-2, -2},				{0, -2},			{2, -2},
{-1, -1},	{0, -1},	{1, -1},
{-2, 0},	{-1, 0},				{1, 0},	{2, 0},
{-1, 1},	{0, 1},		{1, 1},
{-2, 2},				{0, 2},				{2, 2}};
const int coef[16] = {	1,		1,		1,
2,	2,	2,
1,	2,		2,	1,
2,	2,	2,
1,		1,		1};
const int num = 24;
const int yt = blockIdx.y * blockDim.y + threadIdx.y;
const int xt = blockIdx.x * blockDim.x + threadIdx.x;
const int curt = wt * yt + xt;
if (yt < ht && xt < wt && mask[curt] > 127.0f){
float sum[3] = {0}, bsum[3] = {0};
for (int i=0; i<16; i++){
int dxt = xt + dir[i][0];
int dyt = yt + dir[i][1];
int dcurt = wt * dyt + dxt;
int dxb = ox + dxt;
int dyb = oy + dyt;
if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht){
sum[0] += target[dcurt*3 + 0] * coef[i];
sum[1] += target[dcurt*3 + 1] * coef[i];
sum[2] += target[dcurt*3 + 2] * coef[i];
}
else {
sum[0] += target[curt*3 + 0] * coef[i];
sum[1] += target[curt*3 + 1] * coef[i];
sum[2] += target[curt*3 + 2] * coef[i];
}

if (dxt < 0 || dxt >= wt || dyt < 0 || dyt >= ht || mask[dcurt] < 127.0f){
dxb =	dxb <  0 ? 0:
dxb >= wb? wb-1: dxb;
dyb =	dyb <  0 ? 0:
dyb >= hb? hb-1: dyb;
int dcurb = wb * dyb + dxb;
bsum[0] += background[dcurb*3 + 0] * coef[i];
bsum[1] += background[dcurb*3 + 1] * coef[i];
bsum[2] += background[dcurb*3 + 2] * coef[i];
}
}
fixed[curt*3+0] = target[curt*3+0] - sum[0] / num + bsum[0] / num;
fixed[curt*3+1] = target[curt*3+1] - sum[1] / num + bsum[1] / num;
fixed[curt*3+2] = target[curt*3+2] - sum[2] / num + bsum[2] / num;
}
}