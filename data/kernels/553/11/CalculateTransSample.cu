#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateTransSample( const float *input, float *output, const int wtss, const int htss, const int wts, const int hts, const int ratio ){
const int yts = blockIdx.y * blockDim.y + threadIdx.y;
const int xts = blockIdx.x * blockDim.x + threadIdx.x;
const int curst = wts * yts + xts;

const int yt = yts * ratio, xt = xts * ratio;

if (yts < hts && xts < wts){
for (int i=0; i<ratio; i++){
for (int j=0; j<ratio; j++){
if (yt + i < htss && xt + j < wtss){
const int curt = wtss * (yt+i) + (xt+j);
output[curt*3+0] = input[curst*3+0];
output[curt*3+1] = input[curst*3+1];
output[curt*3+2] = input[curst*3+2];
}
}
}
}
}