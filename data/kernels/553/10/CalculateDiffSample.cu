#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateDiffSample( float *cur, float *pre, const int wts, const int hts ){
const int yts = blockIdx.y * blockDim.y + threadIdx.y;
const int xts = blockIdx.x * blockDim.x + threadIdx.x;
const int curst = wts * yts + xts;

if (yts < hts && xts < wts){
cur[curst*3+0] -= pre[curst*3+0];
cur[curst*3+1] -= pre[curst*3+1];
cur[curst*3+2] -= pre[curst*3+2];
pre[curst*3+0] = 0;
pre[curst*3+1] = 0;
pre[curst*3+2] = 0;
}
}