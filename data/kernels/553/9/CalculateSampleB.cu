#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateSampleB( const float *background, float *subBG, const int wb, const int hb, const int ws, const int hs, const int sRate ){
const int ys = blockIdx.y * blockDim.y + threadIdx.y;
const int xs = blockIdx.x * blockDim.x + threadIdx.x;
const int curst = ws * ys + xs;

if (ys < hs && xs < ws){
const int yb = ys * sRate;
const int xb = xs * sRate;
int num = 0;
float sum[3] = {0};

for (int i=0; i<sRate; i++){
for (int j=0; j<sRate; j++){
if (yb + i < hb && xb + j < wb){
int curb = wb * (yb+i) + (xb+j);
sum[0] += background[curb*3+0];
sum[1] += background[curb*3+1];
sum[2] += background[curb*3+2];
num++;
}
}
}
subBG[curst*3+0] = sum[0] / num;
subBG[curst*3+1] = sum[1] / num;
subBG[curst*3+2] = sum[2] / num;
}
}