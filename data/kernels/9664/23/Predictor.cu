#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Predictor (const double TIME, double4 *p_pred, float4  *v_pred, float4  *a_pred, double4 *p_corr, double4 *v_corr, double  *loc_time, double4 *acc, double4 *acc1, double4 *acc2, double4 *acc3, int istart, int* nvec, int ppgpus, unsigned int N){



int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
int cost = ppgpus+istart;

if(i>=cost){
i = nvec[i - cost];
if(i>=istart && i < cost)
i=-1;
}
if(i<0)
return;

double timestep = TIME - loc_time[i];
double t2 = timestep * timestep;
double t3 = t2 * timestep;
double t4 = t2 * t2;
double t5 = t4 * timestep;

t2 *= 0.5;
t3 *= 0.1666666666666666666666;
t4 *= 0.0416666666666666666666;
t5 *= 0.0083333333333333333333;

double4 myppred;
myppred.x = p_pred[i].x;
myppred.y = p_pred[i].y;
myppred.z = p_pred[i].z;

float4  mypred;
mypred.x = v_pred[i].x;
mypred.y = v_pred[i].y;
mypred.z = v_pred[i].z;

double4 mypcorr;
mypcorr.x = p_corr[i].x;
mypcorr.y = p_corr[i].y;
mypcorr.z = p_corr[i].z;

double4 myvcorr;
myvcorr.x = v_corr[i].x;
myvcorr.y = v_corr[i].y;
myvcorr.z = v_corr[i].z;

double4 myacc;
myacc.x = acc[i].x;
myacc.y = acc[i].y;
myacc.z = acc[i].z;

double4 myacc1;
myacc1.x = acc1[i].x;
myacc1.y = acc1[i].y;
myacc1.z = acc1[i].z;

double4 myacc2;
myacc2.x = acc2[i].x;
myacc2.y = acc2[i].y;
myacc2.z = acc2[i].z;

double4 myacc3;
myacc3.x = acc3[i].x;
myacc3.y = acc3[i].y;
myacc3.z = acc3[i].z;


myppred.x = mypcorr.x + timestep * myvcorr.x +
t2 * myacc.x  +
t3 * myacc1.x +
t4 * myacc2.x +
t5 * myacc3.x ;

myppred.y = mypcorr.y + timestep * myvcorr.y +
t2 * myacc.y  +
t3 * myacc1.y +
t4 * myacc2.y +
t5 * myacc3.y ;

myppred.z = mypcorr.z + timestep * myvcorr.z +
t2 * myacc.z  +
t3 * myacc1.z +
t4 * myacc2.z +
t5 * myacc3.z ;

p_pred[i].x = myppred.x;
p_pred[i].y = myppred.y;
p_pred[i].z = myppred.z;

mypred.x = myvcorr.x + timestep * myacc.x +
t2 * myacc1.x +
t3 * myacc2.x +
t4 * myacc3.x ;

mypred.y = myvcorr.y + timestep * myacc.y +
t2 * myacc1.y +
t3 * myacc2.y +
t4 * myacc3.y ;

mypred.z = myvcorr.z + timestep * myacc.z +
t2 * myacc1.z +
t3 * myacc2.z +
t4 * myacc3.z ;

v_pred[i].x = mypred.x;
v_pred[i].y = mypred.y;
v_pred[i].z = mypred.z;

mypred.x = myacc.x + timestep * myacc1.x +
t2 * myacc2.x +
t3 * myacc3.x ;

mypred.y = myacc.y + timestep * myacc1.y +
t2 * myacc2.y +
t3 * myacc3.y ;

mypred.z = myacc.z + timestep * myacc1.z +
t2 * myacc2.z +
t3 * myacc3.z ;

a_pred[i].x = mypred.x;
a_pred[i].y = mypred.y;
a_pred[i].z = mypred.z;
}