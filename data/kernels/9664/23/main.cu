#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Predictor.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const double TIME = 1;
double4 *p_pred = NULL;
hipMalloc(&p_pred, XSIZE*YSIZE);
float4 *v_pred = NULL;
hipMalloc(&v_pred, XSIZE*YSIZE);
float4 *a_pred = NULL;
hipMalloc(&a_pred, XSIZE*YSIZE);
double4 *p_corr = NULL;
hipMalloc(&p_corr, XSIZE*YSIZE);
double4 *v_corr = NULL;
hipMalloc(&v_corr, XSIZE*YSIZE);
double *loc_time = NULL;
hipMalloc(&loc_time, XSIZE*YSIZE);
double4 *acc = NULL;
hipMalloc(&acc, XSIZE*YSIZE);
double4 *acc1 = NULL;
hipMalloc(&acc1, XSIZE*YSIZE);
double4 *acc2 = NULL;
hipMalloc(&acc2, XSIZE*YSIZE);
double4 *acc3 = NULL;
hipMalloc(&acc3, XSIZE*YSIZE);
int istart = 1;
int *nvec = NULL;
hipMalloc(&nvec, XSIZE*YSIZE);
int ppgpus = 1;
unsigned int N = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}