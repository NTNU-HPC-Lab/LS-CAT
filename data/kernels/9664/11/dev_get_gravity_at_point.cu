#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dev_get_gravity_at_point( float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *ax, float *ay, float *az, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
float dx, dy, dz, r2, tmp, dr2, eps2_total;
for (int tid=threadIdx.x + blockIdx.x*blockDim.x; tid < n; tid += blockDim.x*gridDim.x){
eps2_total = eps2 + eps[tid]*eps[tid];
ax[tid] = 0;
ay[tid] = 0;
az[tid] = 0;
for (int i=0; i < n_field; i++){
dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
dr2 = dx*dx + dy*dy + dz*dz;
if (dr2 > 0) {
r2 = eps2_total + dr2;
tmp = field_m[i] / (r2 * sqrt(r2));
ax[tid] += tmp * dx;
ay[tid] += tmp * dy;
az[tid] += tmp * dz;
}
}
}
}