#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Reconstruct(int *nex, unsigned long nextsize, double4 *pc, double4 *vc, double4 *a3, double4 *a, double4 *a1, double4 *a2, double4 *pva3, double4 *aaa) {


unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;

int k = gtid/nextsize;
int who = nex[gtid - k*nextsize];

if(gtid<nextsize){
pc[who] = pva3[gtid];
}
else if(gtid >= nextsize && gtid < 2*nextsize){
vc[who] = pva3[gtid];
}
else if(gtid >= 2*nextsize && gtid < 3*nextsize){
a3[who] = pva3[gtid];
}
else if(gtid >= 3*nextsize && gtid < 4*nextsize){
a[who] = aaa[gtid - 3*nextsize];
}
else if(gtid>= 4*nextsize && gtid < 5*nextsize){
a1[who] = aaa[gtid - 3*nextsize];
}
else if(gtid>= 5*nextsize && gtid < 6*nextsize){
a2[who] = aaa[gtid - 3*nextsize];
}


}