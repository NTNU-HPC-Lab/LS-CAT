#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initvectors(double4 *acc3, float4 *apred){
int i = blockIdx.x*blockDim.x + threadIdx.x;
acc3[i].x = acc3[i].y = acc3[i].z = 0.0;
apred[i].x = apred[i].y = apred[i].z = 0.0f;
}