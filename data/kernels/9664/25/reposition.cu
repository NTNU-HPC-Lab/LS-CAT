#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reposition (double4 *ac, double4 *ac1, double4 *ac2, double4 *af, unsigned long nextsize)
{
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

if(i < nextsize){
af[i]              = ac[i];
af[i + nextsize]   = ac1[i];
af[i + 2*nextsize] = ac2[i];
}


}