#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(double4 *ac, double4 *ac1, double4 *ac2, unsigned int bf_real, unsigned int dimension){

unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int k = dimension*bf_real;
double4 myacc;

extern __shared__ double4 shaccelerations[];
double4 *shacc = (double4*) shaccelerations;

if(i < k){
myacc = ac[i];

shacc[threadIdx.x] = ac[i + k];

myacc.x += shacc[threadIdx.x].x;
myacc.y += shacc[threadIdx.x].y;
myacc.z += shacc[threadIdx.x].z;

ac[i] = myacc;
}
else if (i >= k && i < 2*k){
myacc = ac1[i - k];

shacc[threadIdx.x] = ac1[i];

myacc.x += shacc[threadIdx.x].x;
myacc.y += shacc[threadIdx.x].y;
myacc.z += shacc[threadIdx.x].z;

ac1[i - k] = myacc;
}
else {
myacc = ac2[i - 2*k];

shacc[threadIdx.x] = ac2[i - k];

myacc.x += shacc[threadIdx.x].x;
myacc.y += shacc[threadIdx.x].y;
myacc.z += shacc[threadIdx.x].z;

ac2[i - 2*k] = myacc;
}
}