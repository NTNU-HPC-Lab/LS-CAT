#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_potential_gpu(float *m, float *x, float *y, float *z, float *phi, int N, int N1) {
int i,j;
float rijx, rijy, rijz;
float xi, yi, zi;
float potential;
i = threadIdx.x + blockIdx.x*blockDim.x;
if (i < (N1 == 0 ? N : N1))
{
xi = x[i];
yi = y[i];
zi = z[i];

for (j = (N1 == 0 ? 0 : N1); j < N; j++)
{
rijx = xi - x[j];
rijy = yi - y[j];
rijz = zi - z[j];

if (i!=j)
potential -= m[j]/sqrt(rijx*rijx + rijy*rijy + rijz*rijz);
}
phi[i] = potential;
}
}