#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

for (int i = 0; i<num_iterations; i++)
g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}