#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ResetHeap_kernel(int *mplHeap, int *mplHeapPtr, int numBlock)
{
int index = threadIdx.x + blockDim.x * blockIdx.x;
if (index >= numBlock)
return;

if (index == 0)
mplHeapPtr[0] = numBlock - 1;

mplHeap[index] = numBlock - index - 1;
}