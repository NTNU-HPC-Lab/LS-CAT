#include "hip/hip_runtime.h"
#include "includes.h"
/*
* inference-101
*/



#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)



__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];



__device__ uint32_t RGBAPACK_10bit(float red, float green, float blue, uint32_t alpha)
{
uint32_t ARGBpixel = 0;

// Clamp final 10 bit results
red   = min(max(red,   0.0f), 1023.f);
green = min(max(green, 0.0f), 1023.f);
blue  = min(max(blue,  0.0f), 1023.f);

// Convert to 8 bit unsigned integers per color component
ARGBpixel = ((((uint32_t)red   >> 2) << 24) |
(((uint32_t)green >> 2) << 16) |
(((uint32_t)blue  >> 2) <<  8) | (uint32_t)alpha);

return  ARGBpixel;
}
__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{


// Prepare for hue adjustment
/*
float luma, chromaCb, chromaCr;

luma     = (float)yuvi[0];
chromaCb = (float)((int)yuvi[1] - 512.0f);
chromaCr = (float)((int)yuvi[2] - 512.0f);

// Convert YUV To RGB with hue adjustment
*red  = MUL(luma,     constHueColorSpaceMat[0]) +
MUL(chromaCb, constHueColorSpaceMat[1]) +
MUL(chromaCr, constHueColorSpaceMat[2]);
*green= MUL(luma,     constHueColorSpaceMat[3]) +
MUL(chromaCb, constHueColorSpaceMat[4]) +
MUL(chromaCr, constHueColorSpaceMat[5]);
*blue = MUL(luma,     constHueColorSpaceMat[6]) +
MUL(chromaCb, constHueColorSpaceMat[7]) +
MUL(chromaCr, constHueColorSpaceMat[8]);*/

const float luma = float(yuvi[0]);
const float u    = float(yuvi[1]) - 512.0f;
const float v    = float(yuvi[2]) - 512.0f;

/*R = Y + 1.140V
G = Y - 0.395U - 0.581V
B = Y + 2.032U*/

/**green = luma + 1.140f * v;
*blue  = luma - 0.395f * u - 0.581f * v;
*red   = luma + 2.032f * u;*/

*red    = luma + 1.140f * v;
*green  = luma - 0.395f * u - 0.581f * v;
*blue   = luma + 2.032f * u;
}
__global__ void NV12ToARGB(uint32_t *srcImage,     size_t nSourcePitch, uint32_t *dstImage,     size_t nDestPitch, uint32_t width,         uint32_t height)
{
int x, y;
uint32_t yuv101010Pel[2];
uint32_t processingPitch = ((width) + 63) & ~63;
uint32_t dstImagePitch   = nDestPitch >> 2;
uint8_t *srcImageU8     = (uint8_t *)srcImage;

processingPitch = nSourcePitch;

// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
y = blockIdx.y *  blockDim.y       +  threadIdx.y;

if (x >= width)
return; //x = width - 1;

if (y >= height)
return; // y = height - 1;

// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
// if we move to texture we could read 4 luminance values
yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

uint32_t chromaOffset    = processingPitch * height;
int y_chroma = y >> 1;

if (y & 1)  // odd scanline ?
{
uint32_t chromaCb;
uint32_t chromaCr;

chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
{
chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
}

yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
}
else
{
yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
}

// this steps performs the color conversion
uint32_t yuvi[6];
float red[2], green[2], blue[2];

yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

// YUV to RGB Transformation conversion
YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

// Clamp the results to RGBA
dstImage[y * dstImagePitch + x     ] = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}