#include "hip/hip_runtime.h"

#include "cuda_mparticles.cuh"
#include "cuda_bits.h"

// ----------------------------------------------------------------------
// check_in_patch_unordered_slow

template<typename BS>
bool cuda_mparticles<BS>::check_in_patch_unordered_slow()
{
  auto n_prts_by_patch = this->sizeByPatch();

  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->storage.load(off + n), p);
      if (!(bidx >= 0 && bidx <= this->n_blocks)) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_in_patch_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bix_id_unordered_slow
//
// checks that block indices are correct,
// id is just enumerating particles

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_id_unordered_slow()
{
  auto n_prts_by_patch = this->sizeByPatch();

  uint off = 0;
  for (int p = 0; p < this->n_patches(); p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->storage.load(off + n), p);
      if (!(bidx == this->by_block_.d_idx[off+n])) return false;
      if (!(off+n == this->by_block_.d_id[off+n])) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_bidx_id_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_ordered

template<typename BS>
bool cuda_mparticles<BS>::check_ordered()
{
  thrust::host_vector<float4> h_xi4(this->storage.xi4);
  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  thrust::host_vector<uint> h_id(this->by_block_.d_id);

  //printf("check_ordered: need_reorder %s\n", need_reorder ? "true" : "false");

  uint off = 0;
  for (int b = 0; b < this->n_blocks; b++) {
    int p = b / this->n_blocks_per_patch;
    uint off_b = h_off[b], off_e = h_off[b+1];
    if (!(off_e >= off_b)) return false;
    //printf("check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    if (!(off_b == off)) return false;
    for (int n = h_off[b]; n < h_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = h_xi4[h_id[n]];
      } else {
	xi4 = h_xi4[n];
      }
      uint bidx = this->blockIndex(xi4, p);
      //printf("check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("check_ordered: b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	Int3 bpos = this->blockPosition(&xi4.x);
	printf("block_pos %d %d\n", bpos[1], bpos[2]);
      }
      if (!(b == bidx)) return false;
    }
    off += off_e - off_b;
  }
  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_ordered:\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bidx_after_push

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_after_push()
{
  bool ok = true;

  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  thrust::host_vector<uint> h_bidx(this->by_block_.d_idx);
  thrust::host_vector<float4> h_xi4(this->storage.xi4);
  
  for (int p = 0; p < this->n_patches(); p++) {
    int begin = h_off[p * this->n_blocks_per_patch];
    int end = h_off[(p+1) * this->n_blocks_per_patch];
    for (int n = begin; n < end; n++) {
      float4 xi4 = h_xi4[n];
      int bidx = h_bidx[n];
      int bidx2 = this->blockIndex(xi4, p);
      if (bidx2 < 0) bidx2 = this->n_blocks;
      if (bidx != bidx2) {
	mprintf("check_bidx: n %d: xi4 %g %g %g bidx %d/%d\n", n, xi4.x, xi4.y, xi4.z,
		bidx, bidx2);
	ok = false;
      }
    }
  }
  return ok;
}

