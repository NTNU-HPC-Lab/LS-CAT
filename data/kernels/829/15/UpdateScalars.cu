#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateScalars(float *WHAT , float *WITH , float AMOUNT , float *MASS) {
int idx = threadIdx.x + blockIdx.x * blockDim.x; // this defines the element
WHAT[idx] += AMOUNT*WITH[idx]/MASS[idx];
}