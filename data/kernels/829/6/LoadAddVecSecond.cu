#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LoadAddVecSecond(float *vector , float2 *FFT) {

int idx = threadIdx.x + blockIdx.x*blockDim.x; // this should span the full range of the vector
FFT[idx].x *= vector[idx]/sqV;
FFT[idx].y *= vector[idx]/sqV;
}