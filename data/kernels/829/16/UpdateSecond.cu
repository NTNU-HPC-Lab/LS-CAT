#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateSecond(float *WHAT , float *WITH , float AMOUNT , float *MULT)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
WHAT[idx] *=MULT[idx];
WHAT[idx] +=AMOUNT*WITH[idx];
MULT[idx] = 1.0f;
}