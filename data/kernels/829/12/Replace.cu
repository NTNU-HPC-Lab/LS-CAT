#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Replace(float *WHAT , float *WHERE)
{

int idx = threadIdx.x + blockIdx.x*blockDim.x;
WHERE[idx] = WHAT[idx];

}