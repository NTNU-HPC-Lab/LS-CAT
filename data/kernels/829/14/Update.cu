#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Update(float *WHAT , float *WITH , float AMOUNT) {
int idx = threadIdx.x + blockIdx.x * blockDim.x; // which voxel
WHAT[idx] +=AMOUNT*WITH[idx];
}