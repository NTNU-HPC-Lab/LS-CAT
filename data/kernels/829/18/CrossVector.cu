#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CrossVector(float *first , float *second) {

int idx = threadIdx.x + blockIdx.x * blockDim.x; // the element of the vector
first[idx] *= sqrtf(second[idx]);

}