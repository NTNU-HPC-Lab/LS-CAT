#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LoadVec(float *vector , float2 *FFT) {
int idx = threadIdx.x + blockIdx.x*blockDim.x; // this should span the full range of the vector
FFT[idx].x = vector[idx]; // The real part is replaced by the vector value
FFT[idx].y = 0.0f;        // The imaginary part is zero. The following kernel also replaces the imaginary part
}