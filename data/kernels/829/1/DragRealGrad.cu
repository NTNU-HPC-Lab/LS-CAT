#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DragRealGrad(float2 *ORIGIN , float *DEST , float *VEC) {
int idx = threadIdx.x + blockIdx.x*blockDim.x;
DEST[idx] = ORIGIN[idx].x/sqV - VEC[idx];
}