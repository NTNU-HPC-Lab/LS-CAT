#include "hip/hip_runtime.h"
#include "includes.h"



enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void dividKernel(float* c, const float* a, const float* b)
{
int i = threadIdx.x;
c[i] = a[i] / b[i];
}