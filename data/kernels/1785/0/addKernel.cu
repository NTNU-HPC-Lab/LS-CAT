#include "hip/hip_runtime.h"
#include "includes.h"



enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}