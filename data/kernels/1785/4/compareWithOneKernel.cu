#include "hip/hip_runtime.h"
#include "includes.h"



enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void compareWithOneKernel(float* b, const double* a)
{
int i = threadIdx.x;
if(a[i] == 1)
b[i] = b[i] + 1;
}