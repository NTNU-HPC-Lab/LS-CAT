#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void blend_kernel( float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w )
{
int _id = blockIdx.x * blockDim.x + threadIdx.x;
int size = h * w;
if (_id < c * size) {
// _id = dc * size + id
int id = _id % size, dc = _id / size;
// int x = id % w, y = id / w;
float weight = M[id] < 0.05f ? 0.f : alpha;
AP[dc * size + id] =
A[dc * size + id] * weight +
BP[dc * size + id] * (1.f - weight);
}
return ;
}