#include "hip/hip_runtime.h"
#include "includes.h"


// fill an image with a chekcer_board (BGR)
__global__ void replace_image_by_distance_kernel(const unsigned char *pImage, const float* pDepth, const unsigned char *pBackground, unsigned char *result, const float max_value, const unsigned int width, const unsigned int height, const unsigned int image_channels)
{
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if (y >= height || x >= width)
return;

// get the depth of the current pixel
float z_distance = pDepth[y * width + x];

// replace part of the view
int index = (y * width + x) * 3;
if (isfinite(z_distance) && (z_distance > max_value))
{
result[index] = pBackground[index];
result[index + 1] = pBackground[index + 1];
result[index + 2] = pBackground[index + 2];
}
else
{
if (image_channels == 1)//gray image
{
int img_index = y * width + x;
result[index] = pImage[img_index];
result[index + 1] = pImage[img_index];
result[index + 2] = pImage[img_index];
}
else//color image
{
int img_index = (y * width + x) * image_channels;
result[index] = pImage[img_index];
result[index + 1] = pImage[img_index + 1];
result[index + 2] = pImage[img_index + 2];
}
}
}