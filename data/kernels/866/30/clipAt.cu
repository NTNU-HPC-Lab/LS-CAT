#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void clipAt(float* in, float bound, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) {
if (in[tid] > bound) in[tid] = bound;
if (in[tid] < -bound) in[tid] = -bound;
}
}