#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < scalarCount; tid += stride) {
int linearIndex = tid;
int xindex0 = linearIndex / xstrides0;
linearIndex = linearIndex - xstrides0 * xindex0;
int xindex1 = linearIndex / xstrides1;
linearIndex = linearIndex - xstrides1 * xindex1;
int xindex2 = linearIndex / xstrides2;
int xindex3 = linearIndex - xstrides2 * xindex2;
if (xindex3 >= mask[xindex0]) in[tid] = 0;
}
}