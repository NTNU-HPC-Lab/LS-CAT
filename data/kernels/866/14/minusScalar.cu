#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void minusScalar(float* in, float* out, float minus, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] - minus;
}