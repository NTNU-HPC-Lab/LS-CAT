#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < inScalarCount; tid += stride) {
int linearIndex = tid;
int inIndex0 = linearIndex / inStride0;
linearIndex = linearIndex - inIndex0 * inStride0;
int inIndex1 = linearIndex / inStride1;
if (inIndex0 + inIndex1 >= inDim0) return;
out[tid + inIndex1 * inStride0] = in[tid];
}
}