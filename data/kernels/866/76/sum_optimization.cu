#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_optimization(float* in, int inStr0, int inStr1, int inStr2, int inStr3, float* out, int outStr0, int outStr1, int outStr2, int dim, int nElementOut, int dimSize) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (int i = tid; i < nElementOut; i += stride) {
int outOff0 = i / outStr0;
int outOff1temp = i - outOff0 * outStr0;
int outOff1 = outOff1temp / outStr1;
int outOff2 = outOff1temp - outOff1 * outStr1;
for (int j = 0; j < dimSize; j++) {
int inOff;
if (dim == 0) inOff = j * inStr0 + outOff0 * inStr1 + outOff1 * inStr2 + outOff2 * inStr3;
if (dim == 1) inOff = outOff0 * inStr0 + j * inStr1 + outOff1 * inStr2 + outOff2 * inStr3;
if (dim == 2) inOff = outOff0 * inStr0 + outOff1 * inStr1 + j * inStr2 + outOff2 * inStr3;
if (dim == 3) inOff = outOff0 * inStr0 + outOff1 * inStr1 + outOff2 * inStr2 + j * inStr3;
out[i] += in[inOff];
}
}
}