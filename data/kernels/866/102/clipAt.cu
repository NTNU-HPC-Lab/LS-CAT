#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipAt(float* in, float bound, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) {
if (in[tid] > bound) in[tid] = bound;
if (in[tid] < -bound) in[tid] = -bound;
}
}