#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in1[tid] * in2[tid];
}