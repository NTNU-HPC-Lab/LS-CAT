#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divScalar(float* in, float* out, float div, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] / div;
}