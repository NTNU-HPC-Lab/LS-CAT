#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int offset = tid * x_stride + target[tid];
y[tid] = -1 * x[offset];
}