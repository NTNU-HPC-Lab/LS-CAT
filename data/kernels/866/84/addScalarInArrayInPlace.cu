#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addScalarInArrayInPlace(float* in, float* add, float scale, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) in[tid] += add[0] * scale;
}