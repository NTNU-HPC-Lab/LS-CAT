#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "mul_sub_grad.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *in1_x = NULL;
hipMalloc(&in1_x, XSIZE*YSIZE);
float *in1_d = NULL;
hipMalloc(&in1_d, XSIZE*YSIZE);
float *in2_x = NULL;
hipMalloc(&in2_x, XSIZE*YSIZE);
float *in2_d = NULL;
hipMalloc(&in2_d, XSIZE*YSIZE);
float *out = NULL;
hipMalloc(&out, XSIZE*YSIZE);
int in1ScalarCount = 1;
int in2ScalarCount = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
mul_sub_grad<<<gridBlock,threadBlock>>>(in1_x,in1_d,in2_x,in2_d,out,in1ScalarCount,in2ScalarCount);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
mul_sub_grad<<<gridBlock,threadBlock>>>(in1_x,in1_d,in2_x,in2_d,out,in1ScalarCount,in2ScalarCount);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
mul_sub_grad<<<gridBlock,threadBlock>>>(in1_x,in1_d,in2_x,in2_d,out,in1ScalarCount,in2ScalarCount);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}