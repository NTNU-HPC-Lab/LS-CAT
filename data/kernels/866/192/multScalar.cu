#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multScalar(float* in, float* out, float mult, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] * mult;
}