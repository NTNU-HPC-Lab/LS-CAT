#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = log(in[tid]);
}