#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) in_d[tid] += out_d[tid] * out_x[tid];
}