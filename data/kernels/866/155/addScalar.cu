#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addScalar(float* in, float* out, float add, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] + add;
}