#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int stride = gridDim.x * blockDim.x;
for (int i = tid; i < size; i += stride) {
out[i] = in[i] < min_val ? min_val : (in[i] > max_val ? max_val : in[i]);
}
}