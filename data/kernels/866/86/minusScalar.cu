#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minusScalar(float* in, float* out, float minus, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] - minus;
}