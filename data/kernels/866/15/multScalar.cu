#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void multScalar(float* in, float* out, float mult, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] * mult;
}