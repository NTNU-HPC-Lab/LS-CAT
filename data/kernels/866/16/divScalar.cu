#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void divScalar(float* in, float* out, float div, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
for (; tid < size; tid += stride)
if (tid < size) out[tid] = in[tid] / div;
}