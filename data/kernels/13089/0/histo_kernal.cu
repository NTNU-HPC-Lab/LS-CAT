#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void histo_kernal(char *buffer, long size, int *histo )
{
__shared__ int temp[256];
temp[threadIdx.x] = 0;
__syncthreads();

int i = threadIdx.x + blockIdx.x * blockDim.x;
int offset = blockDim.x * gridDim.x;
int z;
while (i < size)
{
z = buffer[i];
atomicAdd( &temp[z], 1);
i += offset;
}
__syncthreads();


atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}