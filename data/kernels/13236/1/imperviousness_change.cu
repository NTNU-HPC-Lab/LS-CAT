#include "hip/hip_runtime.h"
#include "includes.h"
//	INCLUDES

// CUDA

// GIS

/**
* 	PARS
*/
#define 					BLOCK_DIM_small				64
#define 					BLOCK_DIM 					256

static const unsigned int 	threads 					= 512;
bool 						print_intermediate_arrays 	= false;
const char 					*BASE_PATH 					= "/home/giuliano/git/cuda/reduction";

/*
*	kernel labels
*/
const char		*kern_0			= "filter_roi";
const char 		*kern_1 		= "imperviousness_change_histc_sh_4"	;
const char 		*kern_2 		= "imperviousness_change"	;
char			buffer[255];

/*
* 		DEFINE I/O files
*/
// I/–
//const char 		*FIL_ROI 		= "/home/giuliano/git/cuda/reduction/data/ROI.tif";
//const char 		*FIL_BIN1 		= "/home/giuliano/git/cuda/reduction/data/BIN1.tif";
//const char 		*FIL_BIN2 		= "/home/giuliano/git/cuda/reduction/data/BIN2.tif";
const char 		*FIL_ROI 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char 		*FIL_BIN1 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char 		*FIL_BIN2 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

// –/O
const char 		*FIL_LTAKE_grid	= "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char 		*FIL_LTAKE_count= "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

/*	+++++DEFINEs+++++	*/
__global__ void imperviousness_change( const unsigned char *dev_BIN1, const unsigned char *dev_BIN2, unsigned int WIDTH, unsigned int HEIGHT, int *dev_LTAKE_map )
{
unsigned long int x 	= threadIdx.x;
unsigned long int bdx	= blockDim.x;
unsigned long int bix	= blockIdx.x;
unsigned long int tix	= bdx*bix + x;	// offset

if( tix < WIDTH*HEIGHT ){
dev_LTAKE_map[tix]	= (int)((int)dev_BIN2[tix] - (int)dev_BIN1[tix]);
}
}