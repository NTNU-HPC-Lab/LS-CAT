#include "hip/hip_runtime.h"
#include "includes.h"
//	INCLUDES

// CUDA

// GIS

/**
* 	PARS
*/
#define 					BLOCK_DIM_small				64
#define 					BLOCK_DIM 					256

static const unsigned int 	threads 					= 512;
bool 						print_intermediate_arrays 	= false;
const char 					*BASE_PATH 					= "/home/giuliano/git/cuda/reduction";

/*
*	kernel labels
*/
const char		*kern_0			= "filter_roi";
const char 		*kern_1 		= "imperviousness_change_histc_sh_4"	;
const char 		*kern_2 		= "imperviousness_change"	;
char			buffer[255];

/*
* 		DEFINE I/O files
*/
// I/–
//const char 		*FIL_ROI 		= "/home/giuliano/git/cuda/reduction/data/ROI.tif";
//const char 		*FIL_BIN1 		= "/home/giuliano/git/cuda/reduction/data/BIN1.tif";
//const char 		*FIL_BIN2 		= "/home/giuliano/git/cuda/reduction/data/BIN2.tif";
const char 		*FIL_ROI 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char 		*FIL_BIN1 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char 		*FIL_BIN2 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

// –/O
const char 		*FIL_LTAKE_grid	= "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char 		*FIL_LTAKE_count= "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

/*	+++++DEFINEs+++++	*/
__global__ void filter_roi( unsigned char *BIN, const unsigned char *ROI, unsigned int map_len){
unsigned int tid 		= threadIdx.x;
unsigned int bix 		= blockIdx.x;
unsigned int bdx 		= blockDim.x;
unsigned int gdx 		= gridDim.x;
unsigned int i 			= bix*bdx + tid;
unsigned int gridSize 	= bdx*gdx;

while (i < map_len)
{
//BIN[i] *= ROI[i];
BIN[i] = (unsigned char) ((int)BIN[i] * (int)ROI[i]);
i += gridSize;
}
}