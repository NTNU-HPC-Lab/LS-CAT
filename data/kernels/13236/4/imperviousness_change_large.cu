#include "hip/hip_runtime.h"
#include "includes.h"
//	INCLUDES

// CUDA

// GIS

/**
* 	PARS
*/
#define 					BLOCK_DIM_small				64
#define 					BLOCK_DIM 					256

static const unsigned int 	threads 					= 512;
bool 						print_intermediate_arrays 	= false;
const char 					*BASE_PATH 					= "/home/giuliano/git/cuda/reduction";

/*
*	kernel labels
*/
const char		*kern_0			= "filter_roi";
const char 		*kern_1 		= "imperviousness_change_histc_sh_4"	;
const char 		*kern_2 		= "imperviousness_change"	;
char			buffer[255];

/*
* 		DEFINE I/O files
*/
// I/–
//const char 		*FIL_ROI 		= "/home/giuliano/git/cuda/reduction/data/ROI.tif";
//const char 		*FIL_BIN1 		= "/home/giuliano/git/cuda/reduction/data/BIN1.tif";
//const char 		*FIL_BIN2 		= "/home/giuliano/git/cuda/reduction/data/BIN2.tif";
const char 		*FIL_ROI 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char 		*FIL_BIN1 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char 		*FIL_BIN2 		= "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

// –/O
const char 		*FIL_LTAKE_grid	= "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char 		*FIL_LTAKE_count= "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

/*	+++++DEFINEs+++++	*/
__global__ void imperviousness_change_large( const unsigned char *dev_BIN1, const unsigned char *dev_BIN2, unsigned int WIDTH, unsigned int HEIGHT, int *dev_LTAKE_map, int mapel_per_thread )
{
unsigned long int x 	= threadIdx.x;
unsigned long int bdx	= blockDim.x;
unsigned long int bix	= blockIdx.x;
//unsigned long int gdx	= gridDim.x;
unsigned long int tid	= bdx*bix + x;	// offset
unsigned long int tix	= tid * mapel_per_thread;	// offset

//extern __shared__ int sh_diff[];

if( bdx*bix*mapel_per_thread < WIDTH*HEIGHT ){
//sh_diff[tid] = 0; syncthreads();
for(long int ii=0;ii<mapel_per_thread;ii++){
if( tix+ii < WIDTH*HEIGHT ){
//sh_diff[tid]		= (int)((int)dev_BIN2[tix+ii] - (int)dev_BIN1[tix+ii]);
dev_LTAKE_map[tix+ii] = (int)((int)dev_BIN2[tix+ii] - (int)dev_BIN1[tix+ii]);
} //__syncthreads();
//dev_LTAKE_map[tix+ii]	= sh_diff[tid];
}
}
}