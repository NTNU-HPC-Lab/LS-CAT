#include "hip/hip_runtime.h"
#include "includes.h"

#define PI  3.1415926535897932
#define MAXEQNS    10       // maximum number of differential equations in the system

const int itermax10 = 2;    // number of iterations to use for rk10
const int itermax12 = 1;    // number of additional iterations to use for rk12
const int neqns = 2;        // number of differential equations in the system
const double tol = 1.0e-10; // the error tolerance
const double tol10 = tol / 10;
const bool sho = true;      // set sho to true if you want the simple harmonic oscillator results
// set sho to false, if you want the predator - prey results

// the following constants are the 10th order method's coefficients
const double  a0 = 0;
__constant__ double  a1 = 0.11747233803526765;
__constant__ double  a2 = 0.35738424175967745;
__constant__ double  a3 = 0.64261575824032255;
__constant__ double  a4 = 0.88252766196473235;
const double  a5 = 1.0000000000000000;

__constant__ double  b10 = 0.047323231137709573;
__constant__ double  b11 = 0.077952072407795078;
__constant__ double  b12 = -0.010133421269900587;
__constant__ double  b13 = 0.0028864915990617097;
__constant__ double  b14 = -0.00055603583939812082;
__constant__ double  b20 = 0.021779075831486075;
__constant__ double  b21 = 0.22367959757928498;
__constant__ double  b22 = 0.12204792759220492;
__constant__ double  b23 = -0.012091266674498959;
__constant__ double  b24 = 0.0019689074312004371;
__constant__ double  b30 = 0.044887590835180592;
__constant__ double  b31 = 0.15973856856089786;
__constant__ double  b32 = 0.32285378852557547;
__constant__ double  b33 = 0.12204792759220492;
__constant__ double  b34 = -0.0069121172735362915;
__constant__ double  b40 = 0.019343435528957094;
__constant__ double  b41 = 0.22312684732165494;
__constant__ double  b42 = 0.23418268877986459;
__constant__ double  b43 = 0.32792261792646064;
__constant__ double  b44 = 0.077952072407795078;
const double  b50 = 0.066666666666666667;
const double  b51 = 0.10981508874708385;
const double  b52 = 0.37359383699761912;
const double  b53 = 0.18126454003786724;
const double  b54 = 0.26865986755076313;

const double  c0 = 0.033333333333333333;
const double  c1 = 0.18923747814892349;
const double  c2 = 0.27742918851774318;
const double  c3 = 0.27742918851774318;
const double  c4 = 0.18923747814892349;
const double  c5 = 0.033333333333333333;

// the following coefficients allow us to get rk12 internal xk values from rk10 fk values
__constant__ double  g10 = 0.043407276098971173;
__constant__ double  g11 = 0.049891561330903419;
__constant__ double  g12 = -0.012483721919363355;
__constant__ double  g13 = 0.0064848904066894701;
__constant__ double  g14 = -0.0038158693974615597;
__constant__ double  g15 = 0.0014039153409773882;
__constant__ double  g20 = 0.030385164419638569;
__constant__ double  g21 = 0.19605322645426044;
__constant__ double  g22 = 0.047860687574395354;
__constant__ double  g23 = -0.012887249003100515;
__constant__ double  g24 = 0.0064058521980400821;
__constant__ double  g25 = -0.0022420783785910372;
__constant__ double  g30 = 0.032291666666666667;
__constant__ double  g31 = 0.19311806292811784;
__constant__ double  g32 = 0.25797759963091718;
__constant__ double  g33 = 0.019451588886825999;
__constant__ double  g34 = -0.0038805847791943522;
__constant__ double  g35 = 0.0010416666666666667;
__constant__ double  g40 = 0.035575411711924371;
__constant__ double  g41 = 0.18283162595088341;
__constant__ double  g42 = 0.29031643752084369;
__constant__ double  g43 = 0.22956850094334782;
__constant__ double  g44 = -0.0068157483053369507;
__constant__ double  g45 = 0.0029481689136947641;
__constant__ double  g50 = 0.031929417992355945;
__constant__ double  g51 = 0.19305334754638505;
__constant__ double  g52 = 0.27094429811105371;
__constant__ double  g53 = 0.28991291043710653;
__constant__ double  g54 = 0.13934591681802007;
__constant__ double  g55 = -0.010073942765637839;
const double  g60 = 0.033333333333333333;
const double  g61 = 0.18923747814892349;
const double  g62 = 0.27742918851774318;
const double  g63 = 0.27742918851774318;
const double  g64 = 0.18923747814892349;
const double  g65 = 0.033333333333333333;

// the following constants are the 12th order method's coefficients
const double  ah0 = 0.0;
const double  ah1 = 0.084888051860716535;
const double  ah2 = 0.26557560326464289;
const double  ah3 = 0.50000000000000000;
const double  ah4 = 0.73442439673535711;
const double  ah5 = 0.91511194813928346;
const double  ah6 = 1.0000000000000000;

__constant__ double  bh10 = 0.033684534770907752;
__constant__ double  bh11 = 0.057301749935629582;
__constant__ double  bh12 = -0.0082444880936983822;
__constant__ double  bh13 = 0.0029151263642014432;
__constant__ double  bh14 = -0.00096482361331657787;
__constant__ double  bh15 = 0.00019595249699271744;
__constant__ double  bh20 = 0.015902242088596380;
__constant__ double  bh21 = 0.16276437062291593;
__constant__ double  bh22 = 0.096031583397703751;
__constant__ double  bh23 = -0.011758319711158930;
__constant__ double  bh24 = 0.0032543514515832418;
__constant__ double  bh25 = -0.00061862458499748489;
__constant__ double  bh30 = 0.031250000000000000;
__constant__ double  bh31 = 0.11881843285766042;
__constant__ double  bh32 = 0.24868761828096535;
__constant__ double  bh33 = 0.11000000000000000;
__constant__ double  bh34 = -0.010410996557394222;
__constant__ double  bh35 = 0.0016549454187684515;
__constant__ double  bh40 = 0.015902242088596380;
__constant__ double  bh41 = 0.15809680304274781;
__constant__ double  bh42 = 0.18880881534382426;
__constant__ double  bh43 = 0.28087114502765051;
__constant__ double  bh44 = 0.096031583397703751;
__constant__ double  bh45 = -0.0052861921651656089;
__constant__ double  bh50 = 0.033684534770907752;
__constant__ double  bh51 = 0.11440754737426645;
__constant__ double  bh52 = 0.24657204460460206;
__constant__ double  bh53 = 0.20929436236889375;
__constant__ double  bh54 = 0.25385170908498387;
__constant__ double  bh55 = 0.057301749935629582;
const double  bh60 = 0;
const double  bh61 = 0.19581988897471611;
const double  bh62 = 0.14418011102528389;
const double  bh63 = 0.32000000000000000;
const double  bh64 = 0.14418011102528389;
const double  bh65 = 0.19581988897471611;

const double  ch0 = 0.023809523809523810;
const double  ch1 = 0.13841302368078297;
const double  ch2 = 0.21587269060493131;
const double  ch3 = 0.24380952380952381;
const double  ch4 = 0.21587269060493131;
const double  ch5 = 0.13841302368078297;
const double  ch6 = 0.023809523809523810;

__global__ void Order10FkKernel(double*device_X_Total, double* device_X_Not, double* device_F_Not, double h, double*device_f)
{

int tx = threadIdx.x;
device_X_Total[tx] = device_X_Not[tx] + h*((g10*device_F_Not[tx])+ (g11 * device_f[tx]) + (g12 * device_f[tx+2])+ (g13 * device_f[tx + 4]) + (g14 * device_f[tx+ 6])+ (g15 *device_f[tx+8]));
__syncthreads();
device_X_Total[tx+2] = device_X_Not[tx] + h*((g20*device_F_Not[tx])+ (g21 * device_f[tx]) + (g22 * device_f[tx+2])+ (g23 * device_f[tx + 4]) + (g24 * device_f[tx+ 6])+ (g25 *device_f[tx+8]));
__syncthreads();
device_X_Total[tx+4] = device_X_Not[tx] + h*((g30*device_F_Not[tx])+ (g31 * device_f[tx]) + (g32 * device_f[tx+2])+ (g33 * device_f[tx + 4]) + (g34 * device_f[tx+ 6])+ (g35 *device_f[tx+8]));
__syncthreads();
device_X_Total[tx+6] = device_X_Not[tx] + h*((g40*device_F_Not[tx])+ (g41 * device_f[tx]) + (g42 * device_f[tx+2])+ (g43 * device_f[tx + 4]) + (g44 * device_f[tx+ 6])+ (g45 *device_f[tx+8]));
__syncthreads();
device_X_Total[tx+8] = device_X_Not[tx] + h*((g50*device_F_Not[tx])+ (g51 * device_f[tx]) + (g52 * device_f[tx+2])+ (g53 * device_f[tx + 4]) + (g54 * device_f[tx+ 6])+ (g55 *device_f[tx+8]));
__syncthreads();
}