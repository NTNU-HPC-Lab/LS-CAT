#include "hip/hip_runtime.h"
#include "includes.h"
/*
1. Memory Copy Cost   One-Step
2. Straggler: Ring-based
**/
using namespace std;

#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

__global__ void helloFromGPU(void)
{
printf("Hello from GPU\n");
}