#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const int* A, const int* B, int* C, int size)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
for(int n = 0 ; n < 100; n++) {
C[i] += A[i] + B[i];
}
}