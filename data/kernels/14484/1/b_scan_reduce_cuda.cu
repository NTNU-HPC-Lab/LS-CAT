#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void	b_scan_reduce_cuda(int *values, int *cumulative, size_t bins)
{

int		id = blockDim.x * blockIdx.x + threadIdx.x;
int		tid = threadIdx.x;

int		nb_here = (bins - (blockDim.x * blockIdx.x) < blockDim.x) ? bins - (blockDim.x * blockIdx.x) : blockDim.x;

if (tid == 0)
printf("Block %d nb_here = %d\n", blockIdx.x, nb_here);

////////// REDUCE
//
//
int		next_th = 2;
// printf("size = %u\n", size);
for (int threshold = 1; threshold < blockDim.x / 2; threshold = threshold << 1)
{
// printf("Thread %d \t id = %d \t threshold = %d \t NB = 8 \t threadIdx.x = %d \t next_th = %d\n", tid, id, threshold, tid, next_th);
// printf("NB %d \t tid = %d \t next_th = %d\n", (NB - 1), tid, next_th);
if (tid < nb_here && tid - threshold >= 0 && ((nb_here - 1) - tid) % next_th == 0)
{
//	printf("HERE : values[%d] = %d\n", id, values[id] + values[id - threshold]);
values[id] = values[id] + values[id - threshold];
}
next_th = next_th << 1;
__syncthreads();
}

////////// DOWNSWEEP
//
//

values[nb_here - 1] = 0;
next_th = nb_here >> 1;
int	tmp;
for (int threshold = nb_here; threshold > 1; threshold >>= 1)
{
if (tid == 0)
printf("threshold = %d && next_th = %d\n", threshold, next_th);
// printf("threshold = %d\n", threshold);
if (tid < nb_here && tid - next_th >= 0 && (nb_here - 1 - tid) % threshold == 0) {
tmp = values[id];
printf("Thread %d : values[%d] = %d\n", tid, id, tmp + values[id - threshold]);
printf("Thread %d : values[%d] = %d\n", tid, id - threshold, tmp);
values[id] += values[id - next_th];
values[id - next_th] = tmp;
}
next_th = next_th >> 1;
if (tid == 0)
printf("\n");
__syncthreads();
}


// Store into cumulative
cumulative[id] = values[id];
}