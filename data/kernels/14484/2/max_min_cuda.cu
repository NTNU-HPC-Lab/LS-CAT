#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_min_cuda(float *d_in1, float *d_in2, float *d_max, float *d_min, size_t nb)
{
int ft_id = threadIdx.x + blockDim.x * blockIdx.x;
int tid = threadIdx.x;
int size = (blockIdx.x == gridDim.x - 1) ? (nb % blockDim.x) : blockDim.x;

for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
{
if (ft_id + s < nb && tid < s)
{
d_in1[ft_id] = (d_in1[ft_id] > d_in1[ft_id + s]) ? d_in1[ft_id] : d_in1[ft_id + s];
if (size % 2 == 1 && ft_id + s + s == size - 1)
d_in1[ft_id] = (d_in1[ft_id] > d_in1[ft_id + s + s]) ? d_in1[ft_id] : d_in1[ft_id + s + s];
d_in2[ft_id] = (d_in2[ft_id] < d_in2[ft_id + s]) ? d_in2[ft_id] : d_in2[ft_id + s];
if (size % 2 == 1 && ft_id + s + s == size - 1)
d_in2[ft_id] = (d_in2[ft_id] < d_in2[ft_id + s + s]) ? d_in2[ft_id] : d_in2[ft_id + s + s];
}
__syncthreads();
size /= 2;
}
if (tid == 0)
{
d_max[blockIdx.x] = d_in1[ft_id];
d_min[blockIdx.x] = d_in2[ft_id];
}
// __syncthreads();
// for (int i = 0; i < GRID_SIZE; i++)
// 	printf("d_out[%d] = %f\n", i, d_out[i]);
}