#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void	histogram_cuda(int *histogram, float *values, size_t nb, float bin_size, float min, int bins, int nb_thread)
{
// nb = total size of elems


int			id = (blockIdx.x * blockDim.x + threadIdx.x) * ITEMS_PER_THREAD;
int			thread_id = threadIdx.x;
int			*local_hist = (int *)malloc(sizeof(int) * bins);

if (id == 0)
printf("Bin size : %f\n", bin_size);

// Init local histogram
for (int i = 0; i < bins; i++)
local_hist[i] = 0;

// One shared array per bin
extern __shared__ int s_bins[];

// Compute serially local bin
for (int i = 0; i < ITEMS_PER_THREAD; i++)
{
for (int j = 0; j < bins; j += 1)
{
// if (id + i < NB)
// 	printf("values[%d] = %f <= %f\n", id + i, values[id + i], (float)min + (float)(j + 1) * bin_size);

if (id + i < nb && values[id + i] <= ((float)min + (float)(j + 1) * bin_size))
{
local_hist[j] += 1;
//printf("BlockIdx : %d - Thread %d : values[%d] = %f -> local_hist[%d] = %d\n", blockIdx.x, thread_id, id + i, values[id + i], j, local_hist[j]);
break ;
}
}
}
__syncthreads();
// Store local bins into shared bins
for (int i = 0; i < bins; i++)
{
s_bins[THREADS * i + thread_id] = local_hist[i];
//		printf("Block %d - Thread %d : s_bins[%d] = local_hist[%d] = %d\n", blockIdx.x, thread_id, THREADS * i + thread_id, i, local_hist[i]);
}

__syncthreads();

// if (thread_id == 0)
// {
// 	for (int i = 0; i < nb_thread * 3; i++)
// 	{
// 		printf("s_bins[%d] = %d\n", i, s_bins[i]);
// 	}
// }

// Reduce each shared bin
// int size = (blockIdx.x == gridDim.x - 1) ? (NB % blockDim.x) : blockDim.x;

int size = THREADS;

for (size_t s = THREADS / 2; s > 0; s >>= 1)
{
if (thread_id + s < THREADS && thread_id < s)
{
for (size_t j = 0; j < bins; j++)
{
s_bins[j * THREADS + thread_id] = s_bins[j * THREADS + thread_id] + s_bins[j * THREADS + thread_id + s];

if (size % 2 == 1 && thread_id + s + s == size - 1)
s_bins[j * THREADS + thread_id] = s_bins[j * THREADS + thread_id] + s_bins[j * THREADS + thread_id + s + s];
}
}
__syncthreads();
size = s;
}

// Store the result into histogram
if (thread_id == 0)
{
for (int i = 0; i < bins; i++) {
histogram[i + blockIdx.x * bins] = s_bins[THREADS * i];




//		histogram[0 + blockIdx.x * bins] = s_bins[0];
//		histogram[1 + blockIdx.x * bins] = s_bins[THREADS];
//		histogram[2 + blockIdx.x * bins] = s_bins[THREADS * 2];
//		printf("histogram[%d] = %d\n", 0 + blockIdx.x * bins, s_bins[0]);
//		printf("histogram[%d] = %d\n", 1 + blockIdx.x * bins, s_bins[THREADS]);
//  printf("histogram[%d] = %d\n", i + blockIdx.x * bins, s_bins[THREADS * i]);
}
}
}