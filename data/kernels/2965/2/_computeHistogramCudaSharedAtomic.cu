#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

using HistType = uint32_t;

enum class Mode {
CPU,
OMP,
OMP_NOATOMIC,
CUDA,
CUDA_NOATOMIC,
CUDA_SHARED,
};

enum class AtomicTypeCuda {
NONE,
STANDARD,
SHARED,
};

__global__ void _computeHistogramCudaSharedAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {
__shared__ HistType temp[256];
temp[threadIdx.x] = 0;
__syncthreads(); // Zero this block's temporary array

size_t stride = blockDim.x * gridDim.x;

for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < length;
i += stride) {
atomicAdd(&(temp[bytes[i]]), 1u);
// Make a histogram for a fraction of the bytes
}
__syncthreads();

// Now add up the histograms
atomicAdd(&(histogram[threadIdx.x]), temp[threadIdx.x]);

// Lesson: Don't let too many threads touch the same memory addresses at once
}