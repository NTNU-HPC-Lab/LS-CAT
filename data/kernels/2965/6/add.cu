#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(const int *a, const int *b, int *dest, const size_t length) {

for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < length;
tid += blockDim.x * gridDim.x) {
dest[tid] = a[tid] + b[tid];
}
}