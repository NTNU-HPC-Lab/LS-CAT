#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(const int *a, const int *b, int *dest, const size_t length) {
int tid = blockIdx.x;

if (tid < length) {
dest[tid] = a[tid] - b[tid];
}
}