#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

using HistType = uint32_t;

enum class Mode {
CPU,
OMP,
OMP_NOATOMIC,
CUDA,
CUDA_NOATOMIC,
CUDA_SHARED,
};

enum class AtomicTypeCuda {
NONE,
STANDARD,
SHARED,
};

__global__ void _computeHistogramCudaStandardAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {

size_t stride = blockDim.x * gridDim.x;

for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < length;
i += stride) {
atomicAdd(&(histogram[bytes[i]]), 1u);
}
}