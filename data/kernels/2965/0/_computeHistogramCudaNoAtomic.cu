#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;

using HistType = uint32_t;

enum class Mode {
CPU,
OMP,
OMP_NOATOMIC,
CUDA,
CUDA_NOATOMIC,
CUDA_SHARED,
};

enum class AtomicTypeCuda {
NONE,
STANDARD,
SHARED,
};

__global__ void _computeHistogramCudaNoAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {
size_t stride = blockDim.x * gridDim.x;

for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < length;
i += stride) {
histogram[bytes[i]]++;
}
}