#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

/* Utility function, use to do error checking.

Use this function like this:

checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

And to check the result of a kernel invocation:

checkCudaCall(hipGetLastError());
*/
__global__ void vectorTransformKernel(float* A, float* B, float* Result) {
// insert operation here
int i = threadIdx.x + blockDim.x * blockIdx.x;
if(i < 1000000)
Result[i] = Result[i] + A[i] * B[i];
}