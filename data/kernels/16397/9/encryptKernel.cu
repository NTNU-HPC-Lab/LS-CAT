#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int n, char *key, int keySize) {
unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < n)
deviceDataOut[index] = deviceDataIn[index] + key[index % keySize];
}