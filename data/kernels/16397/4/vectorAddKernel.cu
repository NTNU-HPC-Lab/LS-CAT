#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddKernel(float* A, float* B, float* Result) {
// insert operation here
int i = threadIdx.x + blockDim.x * blockIdx.x;
Result[i] = A[i] + B[i];
}