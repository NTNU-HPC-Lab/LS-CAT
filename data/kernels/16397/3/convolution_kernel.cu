#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution_kernel(float *output, float *input, float *filter) {
//declare shared memory for this thread block
//the area reserved is equal to the thread block size plus
//the size of the border needed for the computation

//Write a for loop that loads all values needed by this thread block
//from global memory (input) and stores it into shared memory (sh_input)
//that is local to this thread block
//for ( ... ) {
//for ( ... ) {
//...
//}
//}

//synchronize to make all writes visible to all threads within the thread block

//compute using shared memory

//store result in the global memory

//store result to global memory
}