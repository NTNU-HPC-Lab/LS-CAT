#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int n) {
unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < n)
deviceDataOut[index] = deviceDataIn[index]+1;
}