#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int n, char *key, int keySize) {
unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < n)
deviceDataOut[index] = deviceDataIn[index] - key[index % keySize];
}