#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy( float *v4, const float *v3, const int n ) {
for(int i=blockIdx.x*blockDim.x+threadIdx.x;i<n;i+=blockDim.x*gridDim.x) {
v4[i*8+0] = v3[i*6+0];
v4[i*8+1] = v3[i*6+1];
v4[i*8+2] = v3[i*6+2];
v4[i*8+4] = v3[i*6+3];
v4[i*8+5] = v3[i*6+4];
v4[i*8+6] = v3[i*6+5];
}
}