#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _gather(const float * input, const int * indices, float * output, const int n)
{
const int tid = threadIdx.x + blockDim.x * blockIdx.x;

if (tid < n)
output[tid] = input[(tid % 6) + 6 * indices[tid / 6]];
}