#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "exclscnmb2e.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int *d_data0 = NULL;
hipMalloc(&d_data0, XSIZE*YSIZE);
int *d_output0 = NULL;
hipMalloc(&d_output0, XSIZE*YSIZE);
int *d_data1 = NULL;
hipMalloc(&d_data1, XSIZE*YSIZE);
int *d_output1 = NULL;
hipMalloc(&d_output1, XSIZE*YSIZE);
int *d_data2 = NULL;
hipMalloc(&d_data2, XSIZE*YSIZE);
int *d_output2 = NULL;
hipMalloc(&d_output2, XSIZE*YSIZE);
int *d_data3 = NULL;
hipMalloc(&d_data3, XSIZE*YSIZE);
int *d_output3 = NULL;
hipMalloc(&d_output3, XSIZE*YSIZE);
int *d_data4 = NULL;
hipMalloc(&d_data4, XSIZE*YSIZE);
int *d_output4 = NULL;
hipMalloc(&d_output4, XSIZE*YSIZE);
int *d_data5 = NULL;
hipMalloc(&d_data5, XSIZE*YSIZE);
int *d_output5 = NULL;
hipMalloc(&d_output5, XSIZE*YSIZE);
int *d_data6 = NULL;
hipMalloc(&d_data6, XSIZE*YSIZE);
int *d_output6 = NULL;
hipMalloc(&d_output6, XSIZE*YSIZE);
int *d_data7 = NULL;
hipMalloc(&d_data7, XSIZE*YSIZE);
int *d_output7 = NULL;
hipMalloc(&d_output7, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
exclscnmb2e<<<gridBlock,threadBlock>>>(d_data0,d_output0,d_data1,d_output1,d_data2,d_output2,d_data3,d_output3,d_data4,d_output4,d_data5,d_output5,d_data6,d_output6,d_data7,d_output7);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
exclscnmb2e<<<gridBlock,threadBlock>>>(d_data0,d_output0,d_data1,d_output1,d_data2,d_output2,d_data3,d_output3,d_data4,d_output4,d_data5,d_output5,d_data6,d_output6,d_data7,d_output7);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
exclscnmb2e<<<gridBlock,threadBlock>>>(d_data0,d_output0,d_data1,d_output1,d_data2,d_output2,d_data3,d_output3,d_data4,d_output4,d_data5,d_output5,d_data6,d_output6,d_data7,d_output7);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}