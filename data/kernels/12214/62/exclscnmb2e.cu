#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exclscnmb2e(int *d_data0, int *d_output0, int *d_data1, int *d_output1, int *d_data2, int *d_output2, int *d_data3, int *d_output3, int *d_data4, int *d_output4, int *d_data5, int *d_output5, int *d_data6, int *d_output6, int *d_data7, int *d_output7) {
const int twid=threadIdx.x;
switch(blockIdx.x) {
case 0:
if(twid<2) {
d_output0[twid]=d_data0[0]*twid;
}
return;
case 1:
if(twid<2) {
d_output1[twid]=d_data1[0]*twid;
}
return;
case 2:
if(twid<2) {
d_output2[twid]=d_data2[0]*twid;
}
return;
case 3:
if(twid<2) {
d_output3[twid]=d_data3[0]*twid;
}
return;
case 4:
if(twid<2) {
d_output4[twid]=d_data4[0]*twid;
}
return;
case 5:
if(twid<2) {
d_output5[twid]=d_data5[0]*twid;
}
return;
case 6:
if(twid<2) {
d_output6[twid]=d_data6[0]*twid;
}
return;
case 7:
if(twid<2) {
d_output7[twid]=d_data7[0]*twid;
}
return;
}
}