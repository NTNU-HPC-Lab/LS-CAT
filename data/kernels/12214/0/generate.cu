#include "hip/hip_runtime.h"
#include "includes.h"
/*
*  test.cu
*  Part of CTC/logistic_rng/
*
*  Created and authored by Yu-Hang Tang on 2015-03-20.
*  Copyright 2015. All rights reserved.
*
*  Users are NOT authorized
*  to employ the present software for their own publications
*  before getting a written permission from the author of this file.
*/




__global__ void generate( float *output, float trunk, int n_particle )
{
for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_particle; i += gridDim.x * blockDim.x ) {
for( int j = 0; j < n_particle; j++ ) {
output[ i + j * n_particle ] = logistic<11, float>( trunk, i, j );
}
}
}