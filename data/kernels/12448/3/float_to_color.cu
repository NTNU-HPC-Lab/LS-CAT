#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float_to_color(uchar4 * pixels, float* in){
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x;

float num = in[offset];

pixels[offset].x = (int)(num*255);
pixels[offset].y = (int)(0);
pixels[offset].z = (int)((MAX_TEMP-num) * 255);
pixels[offset].w = 255;
}