#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

long long remaining_N2(int , int ,long long );
long long remaining_N(int , int ,int );
__global__ void ker(float * cormat, float * upper,int n1,int n)
{
long idx = blockDim.x*blockIdx.x+threadIdx.x;
long i = idx%n1;
long j = idx/n1;
if(i<j && i<n1 && j<n)
{
long tmp=i;
tmp*=(i+1);
tmp/=2;
long tmp_2=i;
tmp_2*=n;
tmp_2=tmp_2-tmp;
tmp_2+=j;
tmp_2-=i;


upper[tmp_2-1]=cormat[j*n+i];
}
}