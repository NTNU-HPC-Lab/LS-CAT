#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

long long remaining_N2(int , int ,long long );
long long remaining_N(int , int ,int );
__global__ void ker2(float * cormat, float * upper,int n1,int n,long long upper_size,int N,int i_so_far,long long M1)
{
long long idx = blockDim.x;
idx*=blockIdx.x;
idx+=threadIdx.x;
long i = idx/n;
long j = idx%n;

if(i<j && i<n1 && j<n)// &&i<N &&j<N && idx<(n1*n))
{
long long tmp=i;
tmp*=(i+1);
tmp/=2;
long long tmp_2=i;
tmp_2*=n;
tmp_2=tmp_2-tmp;
tmp_2+=j;
tmp_2-=i;
long long indexi=n1;
indexi*=j;
indexi=indexi+i;
upper[tmp_2-1]=cormat[indexi];
//if((i==39001 &&j == 69999)||(i==1 && j==2))
// printf("\n\n\n thread:  %f ",upper[tmp_2-1]," ",cormat[indexi]);
}

}