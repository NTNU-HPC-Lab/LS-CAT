#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void render_final(float *points3d_polar, float * selection, float * depth_render, int * img,  int * render, int oh, int ow)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int w = gridDim.x * TILE_DIM;
int h = w /2;
int maxsize = oh * ow;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{

int iw = x;
int ih = y + j;


int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + 1)/2 * ow - 0.5);
int ty = round((points3d_polar[(ih * w + iw) * 3 + 2] + 1)/2 * oh - 0.5);

float tx_offset = ((points3d_polar[(ih * w + iw) * 3 + 1] + 1)/2 * ow - 0.5);
float ty_offset = ((points3d_polar[(ih * w + iw) * 3 + 2] + 1)/2 * oh - 0.5);

float tx00 = 0;
float ty00 = 0;

float tx01 = ((points3d_polar[(ih * w + iw + 1) * 3 + 1] + 1)/2 * ow - 0.5) - tx_offset;
float ty01 = ((points3d_polar[(ih * w + iw + 1) * 3 + 2] + 1)/2 * oh - 0.5) - ty_offset;

float tx10 = ((points3d_polar[((ih + 1) * w + iw) * 3 + 1] + 1)/2 * ow - 0.5) - tx_offset;
float ty10 = ((points3d_polar[((ih + 1) * w + iw) * 3 + 2] + 1)/2 * oh - 0.5) - ty_offset;

float tx11 = ((points3d_polar[((ih+1) * w + iw + 1) * 3 + 1] + 1)/2 * ow - 0.5) - tx_offset;
float ty11 = ((points3d_polar[((ih+1) * w + iw + 1) * 3 + 2] + 1)/2 * oh - 0.5) - ty_offset;

float t00 = 0 * (float)tx00 + (float)tx01 * -1.0/3  + (float)tx10 *  2.0/3   + (float)tx11 *  1.0/3;
float t01 = 0 * (float)ty00 + (float)ty01 * -1.0/3  + (float)ty10 *  2.0/3   + (float)ty11 *  1.0/3;
float t10 = 0 * (float)tx00 + (float)tx01 *  2.0/3  + (float)tx10 * -1.0/3   + (float)tx11 *  1.0/3;
float t11 = 0 * (float)ty00 + (float)ty01 *  2.0/3  + (float)ty10 * -1.0/3   + (float)ty11 *  1.0/3;

float det = t00 * t11 - t01 * t10 + 1e-10;

//printf("%f %f %f %f %f\n", t00, t01, t10, t11, det);

float it00, it01, it10, it11;

it00 = t11/det;
it01 = -t01/det;
it10 = -t10/det;
it11 = t00/det;

//printf("inverse %f %f %f %f\n", it00, it01, it10, it11);

int this_depth = (int)(12800/128 * points3d_polar[(ih * w + iw) * 3 + 0]);
int delta00 = (int)(12800/128 * points3d_polar[(ih * w + iw) * 3 + 0]) - (int)(100 * depth_render[(ty * ow + tx)]);
int delta01 = (int)(12800/128 * points3d_polar[(ih * w + iw + 1) * 3 + 0]) - (int)(100 * depth_render[(ty * ow + tx + 1)]);
int delta10 = (int)(12800/128 * points3d_polar[((ih + 1) * w + iw) * 3 + 0]) - (int)(100 * depth_render[((ty+1) * ow + tx)]);
int delta11 = (int)(12800/128 * points3d_polar[((ih+1) * w + iw + 1) * 3 + 0]) - (int)(100 * depth_render[((ty+1) * ow + tx + 1)]);

int mindelta = min(min(delta00, delta01), min(delta10, delta11));
int maxdelta = max(max(delta00, delta01), max(delta10, delta11));

int depth00 = (int)(12800/128 * points3d_polar[(ih * w + iw) * 3 + 0]);
int depth01 = (int)(12800/128 * points3d_polar[(ih * w + iw + 1) * 3 + 0]);
int depth10 = (int)(12800/128 * points3d_polar[((ih+1) * w + iw) * 3 + 0]);
int depth11 = (int)(12800/128 * points3d_polar[((ih+1) * w + iw+1) * 3 + 0]);
int max_depth =  max(max(depth00, depth10), max(depth01, depth11));
int min_depth =  min(min(depth00, depth10), min(depth01, depth11));
int delta_depth = max_depth - min_depth;

int txmin = floor(tx_offset + min(min(tx00, tx11), min(tx01, tx10)));
int txmax = ceil(tx_offset + max(max(tx00, tx11), max(tx01, tx10)));
int tymin = floor(ty_offset + min(min(ty00, ty11), min(ty01, ty10)));
int tymax = ceil(ty_offset + max(max(ty00, ty11), max(ty01, ty10)));

float newx, newy;
int r,g,b;
int itx, ity;

//render[(ty * ow + tx)] = img[ih * w + iw];
//selection[(ty * ow + tx)] = 1.0;

float tolerance = 0.1 * this_depth > 10? 0.1 * this_depth : 10;
float tolerance2 = 0.05 * max_depth > 10? 0.05 * max_depth: 10;

float flank = 0.01;
if ((delta_depth < tolerance2) && (y > 1 * h/8) && (y < (h*7)/8))
if (((mindelta > - tolerance) && (maxdelta <  tolerance)) && (this_depth < 10000)) {
if (((txmax - txmin) * (tymax - tymin) < 1600) && (txmax - txmin < 40) && (tymax - tymin < 40))
{
for (itx = txmin; itx < txmax; itx ++)
for (ity = tymin; ity < tymax; ity ++)
{ if (( 0 <= itx) && (itx < ow) && ( 0 <= ity) && (ity < oh))
{
newx = (itx - tx_offset) * it00 + it10 * (ity - ty_offset);
newy = (itx - tx_offset) * it01 + it11 * (ity - ty_offset);

//printf("%f %f\n", newx, newy);
if ((newx > -flank) && (newx < 1 + flank) && (newy > -flank) && (newy < 1 + flank))
{
if (newx < 0) newx = 0;
if (newy < 0) newy = 0;
if (newx > 1) newx = 1;
if (newy > 1) newy = 1;

r = img[(ih * w + iw)] / (256*256) * (1-newx) * (1-newy) + img[(ih * w + iw + 1)] / (256*256) * (1-newx) * (newy) + img[((ih+1) * w + iw)] / (256*256) * (newx) * (1-newy) + img[((ih+1) * w + iw + 1)] / (256*256) * newx * newy;
g = img[(ih * w + iw)] / 256 % 256 * (1-newx) * (1-newy) + img[(ih * w + iw + 1)] / 256 % 256 * (1-newx) * (newy) + img[((ih+1) * w + iw)] / 256 % 256  * (newx) * (1-newy)  + img[((ih+1) * w + iw + 1)] / 256 % 256 * newx * newy;
b = img[(ih * w + iw)] % 256 * (1-newx) * (1-newy) + img[(ih * w + iw + 1)] % 256 * (1-newx) * (newy) + img[((ih+1) * w + iw)] % 256 * (newx) * (1-newy)  + img[((ih+1) * w + iw + 1)] % 256 * newx * newy ;

if (r > 255) r = 255;
if (g > 255) g = 255;
if (b > 255) b = 255;

if ((ity * ow + itx > 0) && (ity * ow + itx < maxsize)) {
render[(ity * ow + itx)] = r * 256 * 256 + g * 256 + b;
selection[(ity * ow + itx)] = 1.0 / abs(det);
}
}
}
}

}
}

}



}