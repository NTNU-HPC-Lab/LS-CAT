#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void merge_sum(unsigned char * img_all, unsigned char * img, float * selection, float * selection_sum,  int n, int stride)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
int idx = 0;
float weight = 0;
for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {

img[3*((y+j)*width + x)] = 0;
img[3*((y+j)*width + x)+1] = 0;
img[3*((y+j)*width + x)+2] = 0;

for (idx = 0; idx < n; idx ++) {

weight = selection[idx * stride + ((y+j)*width + x)] / selection_sum[((y+j)*width + x)];
//weight = 0.25;
//weight = 0.5;


img[3*((y+j)*width + x)] += (unsigned char) (img_all[idx * stride * 3 + 3*((y+j)*width + x)] * weight);
img[3*((y+j)*width + x)+1] += (unsigned char) (img_all[idx * stride * 3 + 3*((y+j)*width + x) + 1] * weight);
img[3*((y+j)*width + x)+2] += (unsigned char)(img_all[idx * stride * 3 + 3*((y+j)*width + x) + 2] * weight);

}


}
}