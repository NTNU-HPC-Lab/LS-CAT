#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sum_weights(float * selection_sum,  float * selection, int n, int stride) {
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
int idx = 0;
for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
selection_sum[((y+j)*width + x)] = 0;
for ( idx = 0; idx < n; idx ++) {
atomicAdd(&(selection_sum[((y+j)*width + x)]),  selection[idx * stride + ((y+j)*width + x)]);
}
}
}