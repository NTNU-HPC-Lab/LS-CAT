#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to3d_point(float *depth, float *points3d)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int w = gridDim.x * TILE_DIM;
int h = w / 2;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{

int iw = x;
int ih = y + j;
float depth_point = depth[ ih*w + iw ] * 128.0;
float phi = ((float)(ih) + 0.5) / float(h) * M_PI;
float theta = ((float)(iw) + 0.5) / float(w) * 2 * M_PI + M_PI;

points3d[(ih * w + iw) * 4 + 0] = depth_point * sin(phi) * cos(theta);
points3d[(ih * w + iw) * 4 + 1] = depth_point * sin(phi) * sin(theta);
points3d[(ih * w + iw) * 4 + 2] = depth_point * cos(phi);
points3d[(ih * w + iw) * 4 + 3] = 1;

}
}