#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_mem(unsigned char *source, unsigned char *render)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
for (int channel = 0; channel < 3; channel ++ )
render[3*((y+j)*width + x) + channel] = source[3 * ((y+j)*width + x) + channel];
}