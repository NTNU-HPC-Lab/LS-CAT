#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_with_average(unsigned char *img, int * nz, int * average, int scale)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
//int h = width /2;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{
int iw = x;
int ih = y + j;

if ((img[3*(ih*width + iw)] + img[3*(ih*width + iw)+1] + img[3*(ih*width + iw)+2] == 0) && (nz[ih/scale * width + iw/scale] > 0))
{
img[3*(ih*width + iw)] = (unsigned char)(average[3*(ih/scale*width + iw/scale)] / nz[ih/scale * width + iw/scale]);
img[3*(ih*width + iw) + 1] = (unsigned char)(average[3*(ih/scale*width + iw/scale) + 1] / nz[ih/scale * width + iw/scale]);
img[3*(ih*width + iw) + 2] = (unsigned char)(average[3*(ih/scale*width + iw/scale) + 2] / nz[ih/scale * width + iw/scale]);
}

}
}