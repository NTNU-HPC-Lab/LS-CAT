#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void char_to_int(int * img2, unsigned char * img)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
img2[(y+j)*width + x] =  img[3*((y+j)*width + x) + 0] * 256 * 256 + img[3*((y+j)*width + x) + 1] * 256 + img[3*((y+j)*width + x) + 2];
}