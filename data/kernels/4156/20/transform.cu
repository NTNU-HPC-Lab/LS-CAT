#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transform(float *points3d_after, float *points3d, float * transformation_matrix)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int w = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{
int iw = x;
int ih = y + j;
for (int ic = 0; ic < 3; ic ++) {
points3d_after[(ih * w + iw) * 3 + ic] = points3d[(ih * w + iw) * 4 + 0] * transformation_matrix[4 * ic + 0]
+ points3d[(ih * w + iw) * 4 + 1] * transformation_matrix[4 * ic + 1]
+ points3d[(ih * w + iw) * 4 + 2] * transformation_matrix[4 * ic + 2]
+ points3d[(ih * w + iw) * 4 + 3] * transformation_matrix[4 * ic + 3];
}
}
}