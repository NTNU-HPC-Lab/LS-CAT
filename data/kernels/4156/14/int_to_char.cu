#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void int_to_char(int * img2, unsigned char * img)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
img[3*((y+j)*width + x)] = img2[(y+j)*width + x] / (256*256);
img[3*((y+j)*width + x)+1] = img2[(y+j)*width + x] / 256 % 256;
img[3*((y+j)*width + x)+2] = img2[(y+j)*width + x] % 256;
}
}