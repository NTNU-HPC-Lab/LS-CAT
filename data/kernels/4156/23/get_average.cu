#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_average(unsigned char * img, int * nz, int * average, int scale)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
//int h = width /2;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{
int iw = x;
int ih = y + j;

if (img[3*(ih*width + iw)] + img[3*(ih*width + iw)+1] + img[3*(ih*width + iw)+2] > 0)
{
//nz[ih/3 * width + iw/3] += 1;
//average[3*(ih/3*width + iw/3)] += (int)img[3*(ih*width + iw)];
//average[3*(ih/3*width + iw/3)+1] += (int)img[3*(ih*width + iw)+1];
//average[3*(ih/3*width + iw/3)+2] += (int)img[3*(ih*width + iw)+2];

atomicAdd(&(nz[ih/scale * width + iw/scale]), 1);
atomicAdd(&(average[3*(ih/scale*width + iw/scale)]), (int)img[3*(ih*width + iw)]);
atomicAdd(&(average[3*(ih/scale*width + iw/scale)+1]), (int)img[3*(ih*width + iw)+1]);
atomicAdd(&(average[3*(ih/scale*width + iw/scale)+2]), (int)img[3*(ih*width + iw)+2]);

}

}
}