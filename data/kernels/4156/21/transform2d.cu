#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transform2d(float *points3d_after, float fov_scale)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int w = gridDim.x * TILE_DIM;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{
int iw = x;
int ih = y + j;
float x = points3d_after[(ih * w + iw) * 3 + 0];
float y = points3d_after[(ih * w + iw) * 3 + 1];
float z = points3d_after[(ih * w + iw) * 3 + 2];

points3d_after[(ih * w + iw) * 3 + 0] = x;//sqrt(x * x + y * y + z * z);
//points3d_after[(ih * w + iw) * 3 + 1] = atan2(y, x);
//points3d_after[(ih * w + iw) * 3 + 2] = atan2(sqrt(x * x + y * y), z);

float x2 = fov_scale * x;
if ((x2 > 0) && (y < x2 * 1.1) && (y > -x2 * 1.1) && (z < x2 * 1.1) && (z > -x2 * 1.1)) {
points3d_after[(ih * w + iw) * 3 + 1] = y / (x2 + 1e-5);
points3d_after[(ih * w + iw) * 3 + 2] = -z / (x2 + 1e-5);
}
else {
points3d_after[(ih * w + iw) * 3 + 1] = -1;
points3d_after[(ih * w + iw) * 3 + 2] = -1;
}
}
}