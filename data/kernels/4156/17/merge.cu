#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void merge(unsigned char * img_all, unsigned char * img, float * selection, int n, int stride)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int width = gridDim.x * TILE_DIM;
int idx = 0;
float sum = 0;
float weight = 0;
for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
sum = 0;
for (idx = 0; idx < n; idx ++) sum += selection[idx * stride + ((y+j)*width + x)];

for (idx = 0; idx < n; idx ++) selection[idx * stride + ((y+j)*width + x)] /= (sum + 1e-5);

img[3*((y+j)*width + x)] = 0;
img[3*((y+j)*width + x)+1] = 0;
img[3*((y+j)*width + x)+2] = 0;

for (idx = 0; idx < n; idx ++) {

//weight = selection[idx * stride + ((y+j)*width + x)];
weight = 0.25;
//weight = 0.5;


img[3*((y+j)*width + x)] += (unsigned char) (img_all[idx * stride * 3 + 3*((y+j)*width + x)] * weight);
img[3*((y+j)*width + x)+1] += (unsigned char) (img_all[idx * stride * 3 + 3*((y+j)*width + x) + 1] * weight);
img[3*((y+j)*width + x)+2] += (unsigned char)(img_all[idx * stride * 3 + 3*((y+j)*width + x) + 2] * weight);

}


}
}