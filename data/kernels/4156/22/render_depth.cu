#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void render_depth(float *points3d_polar, unsigned int * depth_render)
{
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;
int w = gridDim.x * TILE_DIM;
int h = w /2;

for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
{
int iw = x;
int ih = y + j;
int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
int this_depth = (int)(512 * points3d_polar[(ih * w + iw) * 3 + 0]);
atomicMin(&depth_render[(ty * w + tx)] , this_depth);
}
}