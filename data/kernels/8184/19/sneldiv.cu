#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sneldiv(unsigned short *inA, float *inB, int   *sub, int Nprj, int snno)
{
int idz = threadIdx.x + blockDim.x*blockIdx.x;
if (blockIdx.y<Nprj && idz<snno) {
// inB > only active bins of the subset
// inA > all sinogram bins
float a = (float)inA[snno*sub[blockIdx.y] + idz];
a /= inB[snno*blockIdx.y + idz];//sub[blockIdx.y]
inB[snno*blockIdx.y + idz] = a; //sub[blockIdx.y]
}
}