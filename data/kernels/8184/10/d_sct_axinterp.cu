#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_sct_axinterp(float *sct3d, const float *scts1, const int4 *sctaxR, const float4 *sctaxW, const short *sn1_sn11, int NBIN, int NSN1, int SPN, int offtof)
{
//scatter crystal index
char ics = threadIdx.x;

//unscattered crystal index
char icu = 2 * threadIdx.y;

//span-1 sino index
short sni = blockIdx.x;

float tmp1, tmp2;

tmp1 = sctaxW[sni].x * scts1[NBIN*sctaxR[sni].x + icu*blockDim.x + ics] +
sctaxW[sni].y * scts1[NBIN*sctaxR[sni].y + icu*blockDim.x + ics] +
sctaxW[sni].z * scts1[NBIN*sctaxR[sni].z + icu*blockDim.x + ics] +
sctaxW[sni].w * scts1[NBIN*sctaxR[sni].w + icu*blockDim.x + ics];

//for the rest of the unscattered crystals (due to limited indexing of 1024 in a block)
icu += 1;
tmp2 = sctaxW[sni].x * scts1[NBIN*sctaxR[sni].x + icu*blockDim.x + ics] +
sctaxW[sni].y * scts1[NBIN*sctaxR[sni].y + icu*blockDim.x + ics] +
sctaxW[sni].z * scts1[NBIN*sctaxR[sni].z + icu*blockDim.x + ics] +
sctaxW[sni].w * scts1[NBIN*sctaxR[sni].w + icu*blockDim.x + ics];


//span-1 or span-11 scatter pre-sinogram interpolation
if (SPN == 1) {
sct3d[offtof + sni*NBIN + (icu - 1)*blockDim.x + ics] = tmp1;
sct3d[offtof + sni*NBIN + icu*blockDim.x + ics] = tmp2;
}
else if (SPN == 11) {
//only converting to span-11 when MRD<=60
if (sni<NSN1) {
short sni11 = sn1_sn11[sni];
atomicAdd(sct3d + offtof + sni11*NBIN + (icu - 1)*blockDim.x + ics, tmp1);
atomicAdd(sct3d + offtof + sni11*NBIN + icu*blockDim.x + ics, tmp2);
}
}

}