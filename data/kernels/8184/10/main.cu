#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "d_sct_axinterp.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *sct3d = NULL;
hipMalloc(&sct3d, XSIZE*YSIZE);
const float *scts1 = NULL;
hipMalloc(&scts1, XSIZE*YSIZE);
const int4 *sctaxR = NULL;
hipMalloc(&sctaxR, XSIZE*YSIZE);
const float4 *sctaxW = NULL;
hipMalloc(&sctaxW, XSIZE*YSIZE);
const short *sn1_sn11 = NULL;
hipMalloc(&sn1_sn11, XSIZE*YSIZE);
int NBIN = 1;
int NSN1 = 1;
int SPN = 1;
int offtof = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
d_sct_axinterp<<<gridBlock,threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
d_sct_axinterp<<<gridBlock,threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
d_sct_axinterp<<<gridBlock,threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}