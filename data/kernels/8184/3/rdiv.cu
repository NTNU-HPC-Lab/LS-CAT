#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rdiv(float * res, const unsigned int * fsum, const float * csum) {

int idx = threadIdx.x + blockIdx.x*blockDim.x;
res[idx] = (float)fsum[idx] / csum[idx];
}