#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void var(float * M1, float * M2, float * X, int b, size_t nele) {
int idx = blockIdx.x*blockDim.x + threadIdx.x;
if (idx<nele) {
float delta = X[idx] - M1[idx];
M1[idx] += delta / (b + 1);
M2[idx] += delta*(X[idx] - M1[idx]);
}
}