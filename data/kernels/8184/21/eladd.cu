#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void eladd(float * inA, float * inB, int length)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;
if (idx<length)    inA[idx] += inB[idx];
}