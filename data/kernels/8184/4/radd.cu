#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void radd(float * resp, const float * res, float alpha) {

int idx = threadIdx.x + blockIdx.x*blockDim.x;

resp[idx] = (1 - alpha)*resp[idx] + alpha*res[idx];
}