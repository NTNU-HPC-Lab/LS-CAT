#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elmult(float * inA, float * inB, int length)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;
if (idx<length) inA[idx] *= inB[idx];
}