#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sneladd(float * inA, float * inB, int *sub, int Nprj, int snno)
{
int idz = threadIdx.x + blockDim.x*blockIdx.x;
if (blockIdx.y<Nprj && idz<snno)
inA[snno*blockIdx.y + idz] += inB[snno*sub[blockIdx.y] + idz];//sub[blockIdx.y]
}