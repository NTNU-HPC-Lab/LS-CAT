#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rinit(float * init, const unsigned int * fsum, const float * ncrs) {

int idx = threadIdx.x + blockIdx.x*blockDim.x;
init[idx] = sqrtf((float)fsum[idx] / ncrs[idx]);
}