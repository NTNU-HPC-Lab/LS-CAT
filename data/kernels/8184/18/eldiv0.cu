#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void eldiv0(float * inA, float * inB, int length)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;
if (idx<length)  inA[idx] /= inB[idx];
}