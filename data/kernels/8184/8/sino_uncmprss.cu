#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sino_uncmprss(unsigned int * dsino, unsigned char * p1sino, unsigned char * d1sino, int ifrm, int nele)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
if (idx<nele) {
d1sino[2 * idx] = (unsigned char)((dsino[ifrm*nele + idx] >> 8) & 0x000000ff);
d1sino[2 * idx + 1] = (unsigned char)((dsino[ifrm*nele + idx] >> 24) & 0x000000ff);

p1sino[2 * idx] = (unsigned char)(dsino[ifrm*nele + idx] & 0x000000ff);
p1sino[2 * idx + 1] = (unsigned char)((dsino[ifrm*nele + idx] >> 16) & 0x000000ff);
}
}