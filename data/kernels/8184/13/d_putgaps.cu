#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_putgaps(float *sne7, float *snaw, int *aw2ali, const int snno)
{
//sino index
int sni = threadIdx.x + blockIdx.y*blockDim.x;

//sino bin index
int awi = blockIdx.x;

if (sni<snno) {
sne7[aw2ali[awi] * snno + sni] = snaw[awi*snno + sni];
}
}