#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void elmsk(float *inA, float *inB, bool  *msk, int length)
{
int idx = threadIdx.x + blockDim.x*blockIdx.x;

if (idx<length) {
if (msk[idx]>0) inA[idx] *= inB[idx];
else  inA[idx] = 0;
}
}