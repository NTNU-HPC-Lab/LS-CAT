#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void r_step( float4 *__restrict__ devPos, float4 *__restrict__ deviceVel, unsigned int numBodies, float dt)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index > numBodies) {return;};
devPos[index].x += deviceVel[index].x * dt;
devPos[index].y += deviceVel[index].y * dt;
devPos[index].z += deviceVel[index].z * dt;
}