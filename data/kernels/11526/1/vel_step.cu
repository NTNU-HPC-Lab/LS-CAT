#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vel_step( float4 *__restrict__ deviceVel, float3 *__restrict__ accels, unsigned int numBodies, float dt)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index > numBodies) {return;};
deviceVel[index].x += accels[index].x * 0.5 * dt;
deviceVel[index].y += accels[index].y * 0.5 * dt;
deviceVel[index].z += accels[index].z * 0.5 * dt;
}