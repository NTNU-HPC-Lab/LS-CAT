#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_old( float4 *__restrict__ newPos, float4 *__restrict__ oldPos )
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
oldPos[index] = newPos[index];
}