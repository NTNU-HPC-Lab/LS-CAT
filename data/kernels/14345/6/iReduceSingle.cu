#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iReduceSingle(int *idata, int *single, int ncols) {
int i;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
unsigned int startPos = blockDim.x + threadIdx.x;
int colsPerThread = ncols/blockDim.x;
int myPart = 0;
for(i=0;i<colsPerThread;i++) {
myPart+=idata[startPos+i];
}
sdata[tid]=myPart;
__syncthreads();

unsigned int s;
for(s=1;s<blockDim.x;s*=2){
if(tid%(2*s) == 0){
sdata[tid]+=sdata[tid+s];
}
__syncthreads();
}
if(tid==0)*single=sdata[0];
}