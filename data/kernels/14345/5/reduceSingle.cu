#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSingle(int *idata, int *single, int nrows)
{
// Reduce rows to the first element in each row
int i;
extern __shared__ int parts[];

// Each block gets a row, each thread will reduce part of a row

// Calculate our offset into the row
// The number of cols per thread

// Sum my part of one dimensional array and put it shared memory
parts[threadIdx.x] = 0;
for (i = threadIdx.x; i < nrows; i+=blockDim.x) {
parts[threadIdx.x] += idata[i];
}
int tid = threadIdx.x;
if (tid < 512) { parts[tid] += parts[tid + 512];}
__syncthreads();
if (tid < 256) { parts[tid] += parts[tid + 256];}
__syncthreads();
if (tid < 128) { parts[tid] += parts[tid + 128];}
__syncthreads();
if (tid < 64) { parts[tid] += parts[tid + 64];}
__syncthreads();
if (tid < 32) { parts[tid] += parts[tid + 32];}
__syncthreads();
if(threadIdx.x == 0) {
*single = 0;
for(i = 0; i < 32; i++) {
*single += parts[i];
}
}
}