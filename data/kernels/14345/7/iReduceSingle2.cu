#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iReduceSingle2(int *idata, int *single, unsigned int ncols) {
int i;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
unsigned int startPos = blockDim.x + threadIdx.x;
int colsPerThread = ncols/blockDim.x;
int myPart = 0;
for(i=0;i<colsPerThread;i++) {
myPart+=idata[startPos+i];
}
sdata[tid]=myPart;
__syncthreads();

unsigned int s;
for(s=1;s<blockDim.x;s*=2) {
int index = 2*s*tid;
if(index<blockDim.x) {
sdata[index] += sdata[index+s];
}
__syncthreads();
}
if(tid==0)*single=sdata[0];
}