#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iReduceSum(int *idata, int *odata, unsigned int ncols) {
int i;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
unsigned int startPos = blockDim.x + threadIdx.x;
int colsPerThread = ncols/blockDim.x;
int blockOffset = threadIdx.x *(ncols/blockDim.x);
int myPart = 0;
for(i=0;i<colsPerThread;i++) {
myPart+=idata[blockOffset+startPos+i];
}
sdata[tid]=myPart;
__syncthreads();

unsigned int s;
for(s=1;s<blockDim.x;s*=2){
if(tid%(2*s) == 0){
sdata[tid]+=sdata[tid+s];
}
__syncthreads();
}
if(tid==0)odata[blockIdx.x]=sdata[0];
}