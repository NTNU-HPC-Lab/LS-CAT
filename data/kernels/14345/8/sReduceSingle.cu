#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sReduceSingle(int *idata,int *single,unsigned int ncols) {
int i;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
unsigned int startPos = blockDim.x + threadIdx.x;
int colsPerThread = ncols/blockDim.x;
int myPart = 0;
for(i=0;i<colsPerThread;i++) {
myPart+=idata[startPos+i];
}
sdata[tid]=myPart;
__syncthreads();

unsigned int s;
for(s=blockDim.x/2;s>0;s>>=1) {
if(tid<s) {
sdata[tid] += sdata[tid+s];
}
__syncthreads();
}
if(tid==0)*single=sdata[0];

}