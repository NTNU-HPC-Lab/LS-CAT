#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sReduceSum(int *idata,int *odata,unsigned int ncols) {
int i;
unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];
unsigned int startPos = blockDim.x + threadIdx.x;
int colsPerThread = ncols/blockDim.x;
int blockOffset = threadIdx.x *(ncols/blockDim.x);
int myPart = 0;
for(i=0;i<colsPerThread;i++) {
myPart+=idata[blockOffset+startPos+i];
}
sdata[tid]=myPart;
__syncthreads();

unsigned int s;
for(s=blockDim.x/2;s>0;s>>=1) {
if(tid<s) {
sdata[tid] += sdata[tid+s];
}
__syncthreads();
}
if(tid==0)odata[blockIdx.x]=sdata[0];

}