#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSum(int *idata, int *odata, unsigned int ncols)
{
// Reduce rows to the first element in each row
int i;
int blockOffset;
int rowStartPos;
int colsPerThread;
int *mypart;

// Each block gets a row, each thread will reduce part of a row

// Calculate the offset of the row
blockOffset = blockIdx.x * ncols;
// Calculate our offset into the row
rowStartPos = threadIdx.x * (ncols/blockDim.x);
// The number of cols per thread
colsPerThread = ncols/blockDim.x;

mypart = idata + blockOffset + rowStartPos;

// Sum all of the elements in my thread block and put them
// into the first column spot
for (i = 1; i < colsPerThread; i++) {
mypart[0] += mypart[i];
}
__syncthreads(); // Wait for everyone to complete
// Now reduce all of the threads in my block into the first spot for my row
if(threadIdx.x == 0) {
odata[blockIdx.x] = 0;
for(i = 0; i < blockDim.x; i++) {
odata[blockIdx.x] += mypart[i*colsPerThread];
}
}
// We cant synchronize between blocks, so we will have to start another kernel
}