#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitArrays(float *ip, float *op, float *fp, int *kp, int ncols)
{
int i;
float *fppos, *oppos, *ippos;
int *kppos;
int blockOffset;
int rowStartPos;
int colsPerThread;

// Each block gets a row, each thread will fill part of a row

// Calculate the offset of the row
blockOffset = blockIdx.x * ncols;
// Calculate our offset into the row
rowStartPos = threadIdx.x * (ncols/blockDim.x);
// The number of cols per thread
colsPerThread = ncols/blockDim.x;

ippos = ip + blockOffset+ rowStartPos;
fppos = fp + blockOffset+ rowStartPos;
oppos = op + blockOffset+ rowStartPos;
kppos = kp + blockOffset+ rowStartPos;

for (i = 0; i < colsPerThread; i++) {
fppos[i] = NOTSETLOC; // Not Fixed
ippos[i] = 50;
oppos[i] = 50;
kppos[i] = 1; // Keep Going
}
if(rowStartPos == 0) {
fppos[0] = SETLOC;
ippos[0] = 0;
oppos[0] = 0;
kppos[0] = 0;
}
if(rowStartPos + colsPerThread >= ncols) {
fppos[colsPerThread-1] = SETLOC;
ippos[colsPerThread-1] = 0;
oppos[colsPerThread-1] = 0;
kppos[colsPerThread-1] = 0;
}
if(blockOffset == 0) {
for(i=0;i < colsPerThread; i++) {
fppos[i] = SETLOC;
ippos[i] = 0;
oppos[i] = 0;
kppos[i] = 0;
}
}
if(blockOffset == ncols - 1) {
for(i=0;i < colsPerThread; i++) {
fppos[i] = SETLOC;
ippos[i] = 100;
oppos[i] = 100;
kppos[i] = 0;
}
}
if(blockOffset == 400 && rowStartPos < 330) {
if(rowStartPos + colsPerThread > 330) {
int end = 330 - rowStartPos;
for(i=0;i<end;i++) {
fppos[i] = SETLOC;
ippos[i] = 100;
oppos[i] = 100;
kppos[i] = 0;
}
}
else {
for(i=0;i<colsPerThread;i++) {
fppos[i] = SETLOC;
ippos[i] = 100;
oppos[i] = 100;
kppos[i] = 0;
}
}
}
if(blockOffset == 200 && rowStartPos <= 500 && rowStartPos + colsPerThread >=500) {
i=500-rowStartPos;
fppos[i] = SETLOC;
ippos[i] = 100;
oppos[i] = 100;
kppos[i] = 0;

}
// Insert code to set the rest of the boundary and fixed positions
}