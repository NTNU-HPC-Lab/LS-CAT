#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyNaive(float * A, float * B, float * C, int N,int K,int M)
{

int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
int Col = blockDim.x*blockIdx.x + threadIdx.x;

if(Row<N && Col<M)
{
float Cvalue = 0.0;
int k;
for(k=0;k<K;k++)
{
Cvalue += A[Row*K+k] * B[k*M+Col];
}
C[Row*M+Col] = Cvalue;
}
}