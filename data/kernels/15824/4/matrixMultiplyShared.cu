#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
__shared__ float sA[TILE_SIZE][TILE_SIZE];   // Tile size to store elements in shared memory
__shared__ float sB[TILE_SIZE][TILE_SIZE];

int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
int Col = blockDim.x*blockIdx.x + threadIdx.x;
float Cvalue = 0.0;
sA[threadIdx.y][threadIdx.x] = 0.0;
sB[threadIdx.y][threadIdx.x] = 0.0;

for (int k = 0; k < (((numAColumns - 1)/ TILE_SIZE) + 1); k++)
{
if ( (Row < numARows) && (threadIdx.x + (k*TILE_SIZE)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
{
sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*TILE_SIZE)];
}
else
{
sA[threadIdx.y][threadIdx.x] = 0.0;
}
if ( Col < numBColumns && (threadIdx.y + k*TILE_SIZE) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
{
sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*TILE_SIZE)*numBColumns + Col];
}
else
{
sB[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();

for (int j = 0; j < TILE_SIZE; ++j)//Multiplying Elements present in tile
{
Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
}
}
if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
{
C[Row*numCColumns + Col] = Cvalue;
}
}