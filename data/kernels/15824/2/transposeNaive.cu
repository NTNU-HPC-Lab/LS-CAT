#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaive(float *odata, const float *idata,int idata_rows,int idata_cols)
{

int x = blockIdx.x * TILE_SIZE + threadIdx.x;
int y = blockIdx.y * TILE_SIZE + threadIdx.y;
//int width = gridDim.x * TILE_SIZE;

if(y<idata_rows && x<idata_cols)
odata[x*idata_rows+y] = idata[y*idata_cols+x];
}