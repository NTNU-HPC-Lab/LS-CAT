#include "hip/hip_runtime.h"
#include "includes.h"
// Device code for ICP computation
// Currently working only on performing rotation and translation using cuda


#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_



#define TILE_WIDTH 256




















#endif // #ifndef _ICP_KERNEL_H_
__global__ void CalculateTotalError(double * distance_d, int size_data)
{
__shared__ double error_s[2*TILE_WIDTH];

unsigned int t = threadIdx.x;
unsigned int start = 2*blockDim.x*blockIdx.x;

if(start + t < size_data)
error_s[t] = distance_d[start + t];
else
error_s[t] = 0.0f;
if(start + blockDim.x + t < size_data)
error_s[blockDim.x + t] = distance_d[start + blockDim.x + t];
else
error_s[blockDim.x + t] = 0.0f;

for(unsigned int stride = blockDim.x; stride >= 1; stride >>= 1)
{
__syncthreads();
if(t < stride)
error_s[t] += error_s[t + stride];
}

if(t == 0)
distance_d[blockIdx.x] = error_s[t];

}