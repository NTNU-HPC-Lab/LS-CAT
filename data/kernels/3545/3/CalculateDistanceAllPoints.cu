#include "hip/hip_runtime.h"
#include "includes.h"
// Device code for ICP computation
// Currently working only on performing rotation and translation using cuda


#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_



#define TILE_WIDTH 256




















#endif // #ifndef _ICP_KERNEL_H_
__global__ void CalculateDistanceAllPoints(double * data_x_d, double * data_y_d, double * data_z_d, double * transformed_data_x_d, double * transformed_data_y_d, double * transformed_data_z_d, int * index_d, double * distance_d, int size_data)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;

if(i < size_data)
{
int index = index_d[i];
distance_d[i] = sqrt(pow(data_x_d[index] - transformed_data_x_d[i],2) + pow(data_y_d[index] - transformed_data_y_d[i],2) + pow(data_z_d[index] - transformed_data_z_d[i],2));
}
}