#include "hip/hip_runtime.h"
#include "includes.h"
// Device code for ICP computation
// Currently working only on performing rotation and translation using cuda


#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_



#define TILE_WIDTH 256




















#endif // #ifndef _ICP_KERNEL_H_
__global__ void CalculateBestIndex(double * distance_d, int * bin_index_d, int size_data)
{
__shared__ double distance_s[2*TILE_WIDTH];
__shared__ unsigned int bin_smallest_index[2*TILE_WIDTH];
unsigned int t = threadIdx.x;
unsigned int start = 2*blockDim.x*blockIdx.x;

if(start + t < size_data)
{
distance_s[t] = distance_d[start + t];
bin_smallest_index[t] = bin_index_d[start + t];
}
else
{
distance_s[t] = 65535;
bin_smallest_index[t] = 0;
}
if(start + blockDim.x + t < size_data)
{
distance_s[blockDim.x + t] = distance_d[start + blockDim.x + t];
bin_smallest_index[blockDim.x + t] = bin_index_d[start + blockDim.x + t];
}
else
{
distance_s[blockDim.x + t] = 65535;
bin_smallest_index[blockDim.x + t] = 0;
}

for(unsigned int stride = blockDim.x; stride >= 1; stride >>= 1)
{
__syncthreads();
if(t < stride)
if(distance_s[t] > distance_s[stride + t])
{
bin_smallest_index[t] = bin_smallest_index[stride + t];
distance_s[t] = distance_s[stride + t];
}

}

if(t == 0)
{
distance_d[blockIdx.x] = distance_s[t];
bin_index_d[blockIdx.x] = bin_smallest_index[t];
}


}