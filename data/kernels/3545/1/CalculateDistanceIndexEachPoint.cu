#include "hip/hip_runtime.h"
#include "includes.h"
// Device code for ICP computation
// Currently working only on performing rotation and translation using cuda


#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_



#define TILE_WIDTH 256




















#endif // #ifndef _ICP_KERNEL_H_
__global__ void CalculateDistanceIndexEachPoint(double point_x, double point_y, double point_z, double * data_x_d, double * data_y_d, double * data_z_d, int * bin_index_d, double * distance_d, int size_data)
{
int index = blockDim.x*blockIdx.x + threadIdx.x;
if(index < size_data)
{
distance_d[index] = sqrt(pow(data_x_d[index] - point_x,2) + pow(data_y_d[index] - point_y,2) + pow(data_z_d[index] - point_z,2));
bin_index_d[index] = index;
}

}