#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "CalculateDistanceIndexEachPoint.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
double point_x = 1;
double point_y = 1;
double point_z = 1;
double *data_x_d = NULL;
hipMalloc(&data_x_d, XSIZE*YSIZE);
double *data_y_d = NULL;
hipMalloc(&data_y_d, XSIZE*YSIZE);
double *data_z_d = NULL;
hipMalloc(&data_z_d, XSIZE*YSIZE);
int *bin_index_d = NULL;
hipMalloc(&bin_index_d, XSIZE*YSIZE);
double *distance_d = NULL;
hipMalloc(&distance_d, XSIZE*YSIZE);
int size_data = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
CalculateDistanceIndexEachPoint<<<gridBlock,threadBlock>>>(point_x,point_y,point_z,data_x_d,data_y_d,data_z_d,bin_index_d,distance_d,size_data);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
CalculateDistanceIndexEachPoint<<<gridBlock,threadBlock>>>(point_x,point_y,point_z,data_x_d,data_y_d,data_z_d,bin_index_d,distance_d,size_data);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
CalculateDistanceIndexEachPoint<<<gridBlock,threadBlock>>>(point_x,point_y,point_z,data_x_d,data_y_d,data_z_d,bin_index_d,distance_d,size_data);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}