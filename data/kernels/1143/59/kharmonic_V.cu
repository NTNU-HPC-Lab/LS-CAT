#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void kharmonic_V(double *x, double *y, double *z, double* items, double *Ax, double *Ay, double *Az, double *V){

int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
int zid = blockDim.z*blockIdx.z + threadIdx.z;

double V_x = items[3]*(x[xid]+items[6]);
double V_y = items[10]*items[4]*(y[yid]+items[7]);
double V_z = items[11]*items[5]*(z[zid]+items[8]);

V[gid] = 0.5*items[9]*((V_x*V_x + V_y*V_y + V_z*V_z)
+ (Ax[gid]*Ax[gid] + Ay[gid]*Ay[gid] + Az[gid]*Az[gid]));
}