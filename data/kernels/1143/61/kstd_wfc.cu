#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void kstd_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc){

int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
int zid = blockDim.z*blockIdx.z + threadIdx.z;

phi[gid] = -fmod(winding*atan2(y[yid], x[xid]),2*PI);

wfc[gid].x = exp(-(x[xid]*x[xid]/(items[14]*items[14]*items[15]*items[15])
+ y[yid]*y[yid]/(items[14]*items[14]*items[16]*items[16])
+ z[zid]*z[zid]/(items[14]*items[14]*items[17]*items[17])))
* cos(phi[gid]);
wfc[gid].y = -exp(-(x[xid]*x[xid]/(items[14]*items[14]*items[15]*items[15])
+ y[yid]*y[yid]/(items[14]*items[14]*items[16]*items[16])
+ z[zid]*z[zid]/(items[14]*items[14]*items[17]*items[17])))
* sin(phi[gid]);

}