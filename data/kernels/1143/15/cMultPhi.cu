#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void cMultPhi(double2* in1, double* in2, double2* out){
double2 result;
unsigned int gid = getGid3d3d();
result.x = cos(in2[gid])*in1[gid].x - in1[gid].y*sin(in2[gid]);
result.y = in1[gid].x*sin(in2[gid]) + in1[gid].y*cos(in2[gid]);
out[gid] = result;
}