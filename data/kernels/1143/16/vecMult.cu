#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void vecMult(double2 *in, double *factor, double2 *out){
double2 result;
unsigned int gid = getGid3d3d();
result.x = in[gid].x * factor[gid];
result.y = in[gid].y * factor[gid];
out[gid] = result;
}