#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void cMult(double2* in1, double2* in2, double2* out){
unsigned int gid = getGid3d3d();
double2 result;
double2 tin1 = in1[gid];
double2 tin2 = in2[gid];
result.x = (tin1.x*tin2.x - tin1.y*tin2.y);
result.y = (tin1.x*tin2.y + tin1.y*tin2.x);
out[gid] = result;
}