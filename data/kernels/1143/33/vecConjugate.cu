#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void vecConjugate(double2 *in, double2 *out){
double2 result;
unsigned int gid = getGid3d3d();
result.x = in[gid].x;
result.y = -in[gid].y;
out[gid] = result;
}