#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void l2_norm(double *in1, double *in2, double *out){

int gid = getGid3d3d();
out[gid] = sqrt(in1[gid]*in1[gid] + in2[gid]*in2[gid]);
}