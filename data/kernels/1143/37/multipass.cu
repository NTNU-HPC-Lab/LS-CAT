#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getTid3d3d(){
return blockDim.x * ( blockDim.y * ( blockDim.z + ( threadIdx.z * blockDim.y ) )  + threadIdx.y )  + threadIdx.x;
}
__device__ unsigned int getBid3d3d(){
return blockIdx.x + gridDim.x*(blockIdx.y + gridDim.y * blockIdx.z);
}
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__device__ double2 mult(double2 a, double2 b){
return {a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x};
}
__device__ double2 mult(double2 a, double b){
return {a.x*b, a.y*b};
}
__global__ void multipass(double* input, double* output){
unsigned int tid = threadIdx.x + threadIdx.y*blockDim.x
+ threadIdx.z * blockDim.x * blockDim.y;
unsigned int bid = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;

//unsigned int tid = getTid3d3d();
//unsigned int bid = getBid3d3d();
// printf("bid0=%d\n",bid);

unsigned int gid = getGid3d3d();
extern __shared__ double sdatad[];
sdatad[tid] = input[gid];
__syncthreads();

for(int i = blockDim.x>>1; i > 0; i>>=1){
if(tid < i){
sdatad[tid] += sdatad[tid + i];
}
__syncthreads();
}
if(tid==0){
output[bid] = sdatad[0];
}
}