#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double complexMagnitude(double2 in){
return sqrt(in.x*in.x + in.y*in.y);
}
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void complexMagnitude(double2 *in, double *out){
int gid = getGid3d3d();
out[gid] = sqrt(in[gid].x*in[gid].x + in[gid].y*in[gid].y);
}