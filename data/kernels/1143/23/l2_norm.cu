#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void l2_norm(double2 *in1, double2 *in2, double *out){

int gid = getGid3d3d();
out[gid] = sqrt(in1[gid].x*in1[gid].x + in1[gid].y*in1[gid].y
+ in2[gid].x*in2[gid].x + in2[gid].y*in2[gid].y);
}