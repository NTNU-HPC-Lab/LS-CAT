#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 pow(double2 a, int b){
double r = sqrt(a.x*a.x + a.y*a.y);
double theta = atan(a.y / a.x);
return{pow(r,b)*cos(b*theta),pow(r,b)*sin(b*theta)};
}
__global__ void pow_test(double2 *a, int b, double2 *c){
c[0] = pow(a[0],b);
}