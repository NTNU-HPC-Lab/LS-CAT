#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void pSum(double* in1, double* output, int pass){
unsigned int tid = threadIdx.x;
unsigned int bid = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x;// printf("bid0=%d\n",bid);
unsigned int gid = getGid3d3d();
extern __shared__ double sdata2[];
for(int i = blockDim.x>>1; i > 0; i>>=1){
if(tid < blockDim.x>>1){
sdata2[tid] += sdata2[tid + i];
}
__syncthreads();
}
if(tid==0){
output[bid] = sdata2[0];
}
}