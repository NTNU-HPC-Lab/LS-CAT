#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x
+ gridDim.x * gridDim.y * blockIdx.z;
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
+ (threadIdx.y * blockDim.x)
+ (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
return threadId;
}
__global__ void kring_rotation_Az(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A){
int gid = getGid3d3d();
int xid = blockDim.x*blockIdx.x + threadIdx.x;
int yid = blockDim.y*blockIdx.y + threadIdx.y;
double r = sqrt(x[xid]*x[xid] + y[yid]*y[yid]);
A[gid] = r*omega*omegaX;
}