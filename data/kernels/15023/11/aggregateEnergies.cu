#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x), i;

for (i = 1; i < batchSize; i++)
{
if (idx + i * interval < numEnergies)
{
energies[idx] += energies[idx + i * interval];
energies[idx + i * interval] = 0;
}
}
}