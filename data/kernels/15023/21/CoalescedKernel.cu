#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CoalescedKernel(int *x, int *y, int *z, int *sum)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

sum[idx] = 0;
sum[idx] += x[idx] * x[idx];
sum[idx] += y[idx] * y[idx];
sum[idx] += z[idx] * z[idx];
}