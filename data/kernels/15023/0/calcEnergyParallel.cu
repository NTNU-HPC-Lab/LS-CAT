#include "hip/hip_runtime.h"
#include "includes.h"
/*
This script is a mockup of the fuctionality to be parallelized in the Monte Carlo
Simulation. It calculates "energy" among pairs of "atoms" in a system, and compares
serial and parallel performance.

The command line arguments are as follows:
first argument (optional) - integer representing number of atoms
- defaults to 100
- input -1 to run benchmarking suite for
10000 <= N <= 40000 and specified thread
block size
second argument (optional) - integer <= 1024 representing thread block size
- input -1 to run benchmarking suite for
64 <= BS <= 1024 and specified N value
For example, -1 512 will run all N with block size = 512
-1 or -1 -1 will run all N for all block sizes
20000 -1 will run N = 20000 for all block sizes

Each simulation adds a line into RunLog.log with data about the run.
*/

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Parallel

//Given two indices in an array (representing atoms),
//calculate their product (potential energy),
//and store in energies array.
//Serial
__global__ void calcEnergyParallel(int *atoms, int numAtoms, int *energies, int numEnergies)
{
int atom1 = blockIdx.x, atom2 = blockIdx.y * blockDim.x + threadIdx.x,
energyIdx;

if (atom2 < numAtoms && atom2 > atom1)
{
energyIdx = gridDim.x * atom1 + atom2 - (blockIdx.x + 1) * (blockIdx.x + 2) / 2;
energies[energyIdx] = atoms[atom1] * atoms[atom2];
}
}