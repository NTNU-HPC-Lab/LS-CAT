#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void staticReverse(int *d, int n)
{
__shared__ int s[64];
int t = threadIdx.x;
int tr = n-t-1;
s[t] = d[t];
__syncthreads();
d[t] = s[tr];
}