#include "hip/hip_runtime.h"
#include "includes.h"

// First solution with global memory

// Shared memory residual calculation
// Reduction code from CUDA Slides - Mark Harris

__global__ void gpu_Heat (float *u, float *utmp, float *residual,int N) {

// TODO: kernel computation
int sizey = N;
int j = blockIdx.x * blockDim.x + threadIdx.x;
int i = blockIdx.y * blockDim.y + threadIdx.y;
float diff=0.0;
if( i < N-1 && j < N-1 && i > 0 && j > 0) {
utmp[i*sizey+j]= 0.25 *
(u[ i*sizey     + (j-1) ]+  // left
u[ i*sizey     + (j+1) ]+  // right
u[ (i-1)*sizey + j     ]+  // top
u[ (i+1)*sizey + j     ]); // bottom
diff = utmp[i*sizey+j] - u[i*sizey + j];
residual[i*sizey+j] = diff * diff;
}
}