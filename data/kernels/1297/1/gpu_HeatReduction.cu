#include "hip/hip_runtime.h"
#include "includes.h"

// First solution with global memory

// Shared memory residual calculation
// Reduction code from CUDA Slides - Mark Harris

__global__ void gpu_HeatReduction (float *res, float *result) {

extern __shared__ float sdata[];
unsigned int tid = threadIdx.x;
unsigned int index= blockIdx.x*blockDim.x+ threadIdx.x;

sdata[tid] = res[index];
__syncthreads();


// Reduce the shared table to compute the residual

for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}
if (tid == 0)
{
int blockIndex = blockIdx.x;

result[blockIndex] = sdata[tid];



}

}