#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void trace_subm(int j, int k, int *daG, int *dbG, double *AB, double *A){
int l;
for(l=0; l<(*dbG); l++){
*(A+j*(*daG)+k) += *(AB+j*(*dbG)+l+k*(*dbG)+l);
}
}
__global__ void ptrBp(int *daG, int *dbG, double *ABg, double *Ag) {
int k = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
trace_subm(j, k, daG, dbG, ABg, Ag);
}