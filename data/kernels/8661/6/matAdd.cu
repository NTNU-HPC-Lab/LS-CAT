#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matAdd(int *yd, float *Ag, float *Bg, float *Cg) {
// reverse order of array and gpu idx, to gain speed
int k = blockIdx.x*blockDim.x + threadIdx.x;
int j = blockIdx.y*blockDim.y + threadIdx.y;
*(Cg+j*(*yd)+k) = *(Ag+j*(*yd)+k) + *(Bg+j*(*yd)+k);
}