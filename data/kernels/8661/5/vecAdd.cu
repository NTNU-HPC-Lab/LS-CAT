#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(int *xd, float *Ag, float *Bg, float *Cg) {
// this is a kernel, which state the computations the gpu shall do
//int j = threadIdx.x;
int j = blockIdx.x*blockDim.x + threadIdx.x;
*(Cg+j) = *(Ag+j) + *(Bg+j) + (*xd);
}