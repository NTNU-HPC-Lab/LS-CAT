#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
#define FSize 256
//void convolution(int *InputImage,int width,int height,int *filter,int filterWidth,,int padding,int *result);
using namespace std;

__global__ void shareMatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap)
{
extern __shared__ int tileImage[];

int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
int value=0;
int feathreMapwidth=width-filterWidth+1;
int shareWidth=(TILE_WIDTH+filterWidth-1);

tileImage[threadIdx.y*shareWidth+threadIdx.x]=InputImage[Row*width+Col];
if(threadIdx.x<filterWidth-1)
{
tileImage[threadIdx.y*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[Row*width+Col+TILE_WIDTH];
}
if(threadIdx.y<filterWidth-1)
{
tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x]=InputImage[(Row+TILE_HEIGHT)*width+Col];
}
if(threadIdx.x<filterWidth-1 && threadIdx.y<filterWidth-1)
{
tileImage[(threadIdx.y+TILE_HEIGHT)*shareWidth+threadIdx.x+TILE_WIDTH]=InputImage[(Row+TILE_HEIGHT)*width+Col+TILE_WIDTH];
}

__syncthreads();

if(Row*width+Col<width*height)
{
for(int i=0;i<filterWidth;i++)
{
for(int j=0;j<filterWidth;j++)
{
//value+=filter[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
value+=filter[i*filterWidth+j]* tileImage[(threadIdx.y+i)*shareWidth+threadIdx.x+j];
}
}
//printf("%d %d\n",Row*width+Col,value);
featureMap[feathreMapwidth*Row+Col]=value;
}
}