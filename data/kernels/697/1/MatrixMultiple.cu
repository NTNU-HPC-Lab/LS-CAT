#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
#define FSize 256
//void convolution(int *InputImage,int width,int height,int *filter,int filterWidth,,int padding,int *result);
using namespace std;

__global__ void MatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap)
{
/* get global row col */
int Row=blockIdx.y*TILE_HEIGHT+threadIdx.y;
int Col=blockIdx.x*TILE_WIDTH+threadIdx.x;
int value=0;
int feathreMapwidth=width-filterWidth+1;
if(Row*width+Col<width*height)
{
for(int i=0;i<filterWidth;i++)
{
for(int j=0;j<filterWidth;j++)
{
value+=filter[i*filterWidth+j]* InputImage[(Row+i)*width+Col+j];
}
}
//printf("%d %d\n",Row*width+Col,value);

featureMap[feathreMapwidth*Row+Col]=value;
}
//printf("%d %d\n",Row*width+Col,value);
}