#include "hip/hip_runtime.h"
#include "includes.h"

//function declaration
unsigned int getmax(unsigned int *, unsigned int);
//unsigned int getmaxSeq(unsigned int *, unsigned int);

__global__ void getmaxcu(unsigned int* num, int size, int threadCount)
{
__shared__ int localBiggest[32];
if (threadIdx.x==0) {
for (int i = 0; i < 32; i++) {
localBiggest[i] = 0;
}
}
__syncthreads();

int current =  blockIdx.x *blockDim.x + threadIdx.x;   //get current thread ID
int localBiggestCurrent = (current - blockIdx.x *blockDim.x)/32;   //get currentID's warp number
//if current number is bigger than the biggest number so far in the warp, replace it
if ((num[current] > localBiggest[localBiggestCurrent]) && (current < size)) {
localBiggest[localBiggestCurrent] = num[current];
}
__syncthreads();

//using only one thread, loop through all the biggest numbers in each warp
//and return the biggest number out of them all
if (threadIdx.x==0) {
int biggest = localBiggest[0];
for (int i = 1; i < 32; i++) {
if (biggest < localBiggest[i]) {
biggest = localBiggest[i];
}
}
//once found the biggest number in this block, put back into global array
//num with corresponding block number
num[blockIdx.x] = biggest;
}

}