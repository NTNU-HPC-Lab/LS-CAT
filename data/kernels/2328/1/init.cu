#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

// image size
int rows = 1224, cols = 1624;
int imgSize = rows*cols;

// iterations for stereo matching algorithm
int iteration = 1;

// disparity range
int Dmin = 1;
int Dmax = 80;
int Drange = Dmax - Dmin + 1;
//int winRadius = 9;

// device image pointer
float* dLImgPtr_f = NULL;
float* dRImgPtr_f = NULL;
size_t lPitch, rPitch;

// texture memory for stereo image pair <Type, Dim, ReadMode>
texture<float, 2, hipReadModeElementType> lTex;
texture<float, 2, hipReadModeElementType> rTex;

// timing arrays
const int nt = 2;
double start[nt], end[nt];
double random_start[nt], random_end[nt];
double main_start[nt], main_end[nt];


// evaluate window-based disimilarity
__global__ void init(unsigned int seed, hiprandState_t* states, int cols)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

int idx = y*cols+x;
hiprand_init(seed, idx, 0, &states[idx]);
}