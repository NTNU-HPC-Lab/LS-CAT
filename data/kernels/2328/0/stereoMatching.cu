#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

// image size
int rows = 1224, cols = 1624;
int imgSize = rows*cols;

// iterations for stereo matching algorithm
int iteration = 1;

// disparity range
int Dmin = 1;
int Dmax = 80;
int Drange = Dmax - Dmin + 1;
//int winRadius = 9;

// device image pointer
float* dLImgPtr_f = NULL;
float* dRImgPtr_f = NULL;
size_t lPitch, rPitch;

// texture memory for stereo image pair <Type, Dim, ReadMode>
texture<float, 2, hipReadModeElementType> lTex;
texture<float, 2, hipReadModeElementType> rTex;

// timing arrays
const int nt = 2;
double start[nt], end[nt];
double random_start[nt], random_end[nt];
double main_start[nt], main_end[nt];


// evaluate window-based disimilarity
__device__ float evaluateCost(float u, float v, float matchIdx, int cols, int rows, int winRadius)
{
float cost = 0.0f;

for(int h=-winRadius; h<=winRadius; h++)
{
for(int w=-winRadius; w<=winRadius; w++)
{
cost += fabsf(tex2D(lTex, matchIdx+ w/(float)cols, v+h/(float)rows)
- tex2D(rTex, u+w/(float)cols, v+h/(float)rows));
}
}

return cost;
}
__global__ void stereoMatching(float* dRDispPtr, float* dRPlanes, int cols, int rows, hiprandState* states, int iteration)
{

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;

int winRadius = 9;

// does not need to process borders
if(x>=cols-winRadius || x<winRadius || y>=rows-winRadius || y<winRadius)
return;

float u = x/(float)cols;
float v = y/(float)rows;

int idx = y*cols +x;

// if 1st iteration, enforce planes to be fronto-parallel
if(iteration != 0)
{
// x of a unit normal vector
dRPlanes[idx*3] = 0.0f;
// y
dRPlanes[idx*3+1] = 0.0f;
// z
dRPlanes[idx*3+2] = 1.0f;
}

// evaluate disparity of current pixel
float min_cost = 0.0f;
float cost = 0.0f;
float tmp_disp = dRDispPtr[idx];
float matchIdx = u + tmp_disp*80.0f/(float)cols;

min_cost = evaluateCost(u, v, matchIdx, cols, rows, winRadius);

// evaluate disparity of left neighbor
cost = 0.0f;
tmp_disp = dRDispPtr[idx-1];
matchIdx = u + tmp_disp*80.0f/(float)cols;

cost = evaluateCost(u, v, matchIdx, cols, rows, winRadius);
// update current disparity if lower cost from neighbor's
if(cost < min_cost)
{
min_cost = cost;
dRDispPtr[idx] = tmp_disp;
}

// evaluate disparity of upper neighbor
cost = 0.0f;
tmp_disp =  dRDispPtr[idx-cols];
matchIdx = u + tmp_disp*80.0f/(float)cols;

cost = evaluateCost(u, v, matchIdx, cols, rows, winRadius);

if(cost < min_cost)
{
min_cost = cost;
dRDispPtr[idx] = tmp_disp;
}

// evaluate another valid random disparitiy (within border) in case it is trapped at a local minima
matchIdx= -1.0f;

while(matchIdx <(float)winRadius/cols || matchIdx >=(float)(cols-winRadius)/cols )
{
tmp_disp = hiprand_uniform(&states[idx]);

matchIdx = u + tmp_disp*80.0f/(float)cols;
}

cost = evaluateCost(u, v, matchIdx, cols, rows, winRadius);

if(cost<min_cost)
{
min_cost = cost;
dRDispPtr[idx] = tmp_disp;
}

return;
}