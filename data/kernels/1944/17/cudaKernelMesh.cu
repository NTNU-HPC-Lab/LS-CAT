#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernelMesh(float4* pos, unsigned int width, unsigned int height, float time)
{
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

// calculate uv coordinates
float u = x / (float) width;
float v = y / (float) height;
u = u*2.0f - 1.0f;
v = v*2.0f - 1.0f;

// calculate simple sine wave pattern
float freq = 4.0f;
float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

// write output vertex
pos[y*width+x] = make_float4(u, w, v, __int_as_float(0xff00ff00)); //Color : DirectX ARGB, OpenGL ABGR
}