#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaKernelTexture2D(unsigned char* surface, int width, int height, size_t pitch, float t)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;
unsigned char* pixel;

// in the case where, due to quantization into grids, we have
// more threads than pixels, skip the threads which don't
// correspond to valid pixels
if (x >= width || y >= height) return;

// get a pointer to the pixel at (x,y)
pixel = (unsigned char*)(surface + y*pitch) + 4*x;

// populate it
float value_x = 0.5f + 0.5f*cos(t + 10.0f*( (2.0f*x)/width  - 1.0f ) );
float value_y = 0.5f + 0.5f*cos(t + 10.0f*( (2.0f*y)/height - 1.0f ) );

// Color : DirectX BGRA, OpenGL RGBA
pixel[0] = 255*(0.5f + 0.5f*cos(t));                          // blue
pixel[1] = 255*(0.5*pixel[1]/255.0 + 0.5*pow(value_y, 3.0f)); // green
pixel[2] = 255*(0.5*pixel[0]/255.0 + 0.5*pow(value_x, 3.0f)); // red
pixel[3] = 255;                                               // alpha
}