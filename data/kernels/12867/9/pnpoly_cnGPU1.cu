#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npoint) {
int j, k, c = 0;
for (j = 0, k = nvert-1; j < nvert; k = j++) {
if ( ((vy[j]>py[i]) != (vy[k]>py[i])) &&
(px[i] < (vx[k]-vx[j]) * (py[i]-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
c = !c;
}
cs[i] = c & 1;
}
}