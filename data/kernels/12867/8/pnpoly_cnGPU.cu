#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU(char *cs, const float *px, const float *py, const float *vx, const float *vy, int npoint, int nvert)
{
extern __shared__ int s[];
float *tvx = (float*) s;
float *tvy = (float*)&s[nvert];

int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npoint) {
int j, k, c = 0;
for (j = 0, k = nvert-1; j < nvert; k = j++) {
tvx[j] = vx [j];
tvy[j] = vy [j];
}

__syncthreads();

for (j = 0, k = nvert-1; j < nvert; k = j++) {
if ( ((tvy[j]>py[i]) != (tvy[k]>py[i])) &&
(px[i] < (tvx[k]-tvx[j]) * (py[i]-tvy[j]) / (tvy[k]-tvy[j]) + tvx[j]) )
c = !c;
}
cs[i] = c & 1;
}
}