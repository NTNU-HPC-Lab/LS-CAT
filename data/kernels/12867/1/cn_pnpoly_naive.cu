#include "hip/hip_runtime.h"
#include "includes.h"

#define VERTICES 600

__constant__ float2 d_vertices[VERTICES];
__constant__ float d_slopes[VERTICES];

/*
* This file contains the implementation of a CUDA Kernel for the
* point-in-polygon problem using the crossing number algorithm
*
* The kernel cn_pnpoly is can be tuned using the following parameters:
*    * block_size_x                any sensible thread block size
*    * tile_size                   any sensible tile size value
*    * between_method              any of [0, 1, 2, 3]
*    * use_precomputed_slopes      enable or disable [0, 1]
*    * use_method                  any of [0, 1]
*
* The kernel cn_pnpoly_naive is used for correctness checking.
*
* The algorithm used here is adapted from:
*     'Inclusion of a Point in a Polygon', Dan Sunday, 2001
*     (http://geomalgorithms.com/a03-_inclusion.html)
*
* Author: Ben van Werkhoven <b.vanwerkhoven@esciencecenter.nl>
*/

#ifndef block_size_x
#define block_size_x 256
#endif
#ifndef block_size_y
#define block_size_y 1
#endif
#ifndef block_size_z
#define block_size_z 1
#endif

#ifndef tile_size
#define tile_size 1
#endif



__global__ void cn_pnpoly_naive(int* bitmap, float2* points, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i < n) {
int c = 0;
float2 p = points[i];

int k = VERTICES-1;

for (int j=0; j<VERTICES; k = j++) {    // edge from v to vp
float2 vj = d_vertices[j];
float2 vk = d_vertices[k];

float slope = (vk.x-vj.x) / (vk.y-vj.y);

if ( (  (vj.y>p.y) != (vk.y>p.y)) &&            //if p is between vj and vk vertically
(p.x < slope * (p.y-vj.y) + vj.x) ) {   //if p.x crosses the line vj-vk when moved in positive x-direction
c = !c;
}
}

bitmap[i] = c; // 0 if even (out), and 1 if odd (in)
}


}