#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
__shared__ float tpx;
__shared__ float tpy;
if (i < npoint) {
tpx = px[i];
tpy = py[i];
int j, k, c = 0;
for (j = 0, k = nvert-1; j < nvert; k = j++) {
if ( ((vy[j]>tpy) != (vy[k]>tpy)) &&
(tpx < (vx[k]-vx[j]) * (tpy-vy[j]) / (vy[k]-vy[j]) + vx[j]) )
c = !c;
}
cs[i] = c & 1;
__syncthreads();
}
}