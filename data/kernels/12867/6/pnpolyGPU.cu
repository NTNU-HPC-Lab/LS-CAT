#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpolyGPU(const float *vertex, float testx, float testy, int* results)
{
int id = blockIdx.x;
int indexOriginX = (blockIdx.x + 1) * 3;
int indexOriginY = (blockIdx.x + 1) * 3 + 1;
int indexDestinoX = blockIdx.x * 3;
int indexDestinoY = blockIdx.x * 3 + 1;

if ( ((vertex[indexOriginY]>testy) != (vertex[indexDestinoY]>testy)) && (testx < (vertex[indexDestinoX]-vertex[indexOriginX]) * (testy-vertex[indexOriginY]) / (vertex[indexDestinoY]-vertex[indexOriginY]) + vertex[indexOriginX]) )
results[id] = 1;
else
results[id] = 0;
}