#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
__shared__ float tvx[607];
__shared__ float tvy[607];

int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i < npoint) {
int j, k, c = 0;
for (j = 0, k = nvert-1; j < nvert; k = j++) {
tvx[j] = vx [j];
tvy[j] = vy [j];
if ( ((tvy[j]>py[i]) != (tvy[k]>py[i])) &&
(px[i] < (tvx[k]-tvx[j]) * (py[i]-tvy[j]) / (tvy[k]-tvy[j]) + tvx[j]) )
c = !c;
}
cs[i] = c & 1;
}
__syncthreads();
}