#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;

if (idx < N)
{
c[idx] = a[idx] + b[idx];
}
}