#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearPolynomProbsImpl( const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs, const int* origFIds) {
if (threadIdx.x < batchSize) {
int polynomId = blockIdx.x;

features += threadIdx.x;
probs += threadIdx.x;

while (polynomId < polynomCount) {
int offset = polynomOffsets[polynomId];
int nextOffset = polynomOffsets[polynomId + 1];
const int depth = nextOffset - offset;
const int origFId = origFIds[polynomId];

bool zeroProb = false;
for (int i = 0; i < depth; ++i) {
if (zeroProb) {
continue;
}

const float c = __ldg(conditions + offset + i);

const int f = __ldg(splits + offset + i);
const float x = __ldg(features + f * batchSize);

if (x <= c) {
zeroProb = true;
}
}

float prob = 0.0f;
if (!zeroProb) {
// TODO we store fID = -1 as our bias column, but it's a hack and we need to get rid of this
if (origFId != -1) {
prob = __ldg(features + origFId * batchSize);
} else {
prob = 1.0f;
}
}

probs[polynomId * batchSize] = prob;
polynomId += gridDim.x;
}
}
}