#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearPolynomForwardImpl( const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {

//out: batch_elem0 dim0, dim1, dimk batch_elem1 dim0 dim1 dimk
//so threads
int polynomId = blockIdx.x;
const int dimId = blockIdx.y;

int tid = threadIdx.x;
if (tid >= batchSize) {
return;
}

float sum = 0;
probs += threadIdx.x;
values += dimId;

while (polynomId < polynomCount) {
const float polynomProb = __ldg(probs + polynomId * batchSize); // includes x
const float v = __ldg(values + polynomId * outputDim);
sum += polynomProb * v;
polynomId += gridDim.x;
}

atomicAdd(out + dimId * batchSize + threadIdx.x, sum);
}