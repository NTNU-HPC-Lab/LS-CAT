#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_loss_kernel(float *reduced_loss, float *predict, float *target, float *workspace, int batch_size, int num_outputs)
{
int batch_idx = blockDim.x * blockIdx.x + threadIdx.x;

extern __shared__ float s_data[];
float loss = 0.f;

// each thread calculate entropy for each data and accumulate to shared memory
for (int c = 0; c < num_outputs; c++)
loss += target[batch_idx * num_outputs + c] * logf(predict[batch_idx * num_outputs + c]);
workspace[batch_idx] = -loss;

// then, we do reduction the result to calculate loss using 1 thread block
if (blockIdx.x > 0) return;

// cumulate workspace data
s_data[threadIdx.x] = 0.f;
for (int i = 0; i < batch_size; i += blockDim.x)
{
s_data[threadIdx.x] += workspace[threadIdx.x + i];
}

__syncthreads();

// reduction
for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
{
if (threadIdx.x + stride < batch_size)
s_data[threadIdx.x] += s_data[threadIdx.x + stride];

__syncthreads();
}

if (threadIdx.x == 0) {
reduced_loss[blockIdx.x] = s_data[0];
}
}