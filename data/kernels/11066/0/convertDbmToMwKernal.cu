#include "hip/hip_runtime.h"
#include "includes.h"


// Kind of lame, but just put static file-level variables here for now.
// Pointer to device results array.
float * dev_result = 0;

// Pointer to device data array.
float * dev_data = 0;

// Size of data/result sets (i.e. number of entries in array).
unsigned int testArraySize = 0;

// GPU function to converts the provided dBm value to mW.
// The power in milliwatts (P(mW)) is equal to 1mW times 10 raised by the
// power in decibel-milliwatts (P(dBm)) divided by 10:
// P(mW) = 1mW * 10 ^ (P(dBm) / 10)
__device__ float convertDbmToMw(const float dBm)
{
return powf(10.0f, dBm / 10.0f);
}
__global__ void convertDbmToMwKernal(float * result, const float * data)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
result[i] = convertDbmToMw(data[i]);
}