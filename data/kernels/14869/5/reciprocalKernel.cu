#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
if (idx < vectorSize)
data[idx] = 1.0/data[idx];
}