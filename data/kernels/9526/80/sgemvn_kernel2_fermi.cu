#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemvn_kernel2_fermi(int n, int m, int n1, float alpha,  float* A, int lda, float *x, float *y)
{
int ind = blockIdx.x*num_threads + threadIdx.x;

A += ind;
x += threadIdx.x;

float res = 0.f;

__shared__ float buff[num_threads];
for(int i=0; i<n1; i += num_threads ){
__syncthreads();
buff[threadIdx.x]  = x[i];

__syncthreads();
#pragma unroll
for(int j=0; j < num_threads ; j++){
res+=A[0]*buff[j];
A+=lda;
}
}
#if 0
__syncthreads();

if (m>n1){
buff[threadIdx.x]  = x[n1];

__syncthreads();
for(int j=0; j<(m-n1); j++){
res += A[0]*buff[j];
A+=lda;
}
}
#endif

if (ind<n)
y[ind] = alpha * res;
}