#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_shmem_doublebuffer ( float* y, float* x, float a, clock_t * timer_vals)
{
volatile __shared__ float sdata_x0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y1 [COMPUTE_THREADS_PER_CTA];
int tid = threadIdx.x ;
unsigned int idx0, idx1;
idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
for (int i=0; i < NUM_ITERS; i+=2) {
__syncthreads();
sdata_x0[tid] = x[idx0];
sdata_y0[tid] = y[idx0];
if (i!=0) {
y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT ;
}
__syncthreads();
sdata_x1[tid] = x[idx1];
sdata_y1[tid] = y[idx1];
y[idx0] = a * sdata_x0[tid] + sdata_y0[tid];
idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT ;
}
__syncthreads();
y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
}