#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s ( float* y, float* x, float a, clock_t * timer_vals)
{
for (int i=0; i < NUM_ITERS/4; i++) {
unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;

float4 * x_as_float4 = (float4 *)x;
float4 * y_as_float4 = (float4 *)y;

float4 tmp1_x, tmp1_y;
tmp1_x = x_as_float4[idx];
tmp1_y = y_as_float4[idx];

float4 result_y;
result_y.x = a * tmp1_x.x + tmp1_y.x;
result_y.y = a * tmp1_x.y + tmp1_y.y;
result_y.z = a * tmp1_x.z + tmp1_y.z;
result_y.w = a * tmp1_x.w + tmp1_y.w;
y_as_float4[idx] = result_y;
}
}