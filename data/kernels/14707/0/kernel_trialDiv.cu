#include "hip/hip_runtime.h"
#include "includes.h"
/*****************************************************************************/
// nvcc -O1 -o bpsw bpsw.cu -lrt -lm


// Assertion to check for errors
__global__ void kernel_trialDiv (long* n, int* r) {
int bx = blockIdx.x;      // ID thread
int tx = threadIdx.x;
int i=0;

// Identify the row and column of the Pd element to work on
long memIndex = bx*TILE_WIDTH+tx;
for (i = 0; i < 256; i++)
{
//		r[memIndex] = ((n[memIndex])%(d_sPrimes[i]) == 0)? (r[memIndex] - 1) : r[memIndex];			//ternary is slower than if statement
if (n[memIndex] % d_sPrimes[i] == 0)
r[memIndex] = r[memIndex] - 1;															//r decreases from 1. Only 1s are prime candidates
}

__syncthreads();
}