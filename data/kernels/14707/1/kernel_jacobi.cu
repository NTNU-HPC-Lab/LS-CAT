#include "hip/hip_runtime.h"
#include "includes.h"
/*****************************************************************************/
// nvcc -O1 -o bpsw bpsw.cu -lrt -lm


// Assertion to check for errors
__global__ void kernel_jacobi(long* nArray, long* dArray, long len) {
int bx = blockIdx.x;      // ID thread
int tx = threadIdx.x;
int result, t;
long d, dAbs, sign, temp, n1, d1;
// Identify the row and column of the Pd element to work on
long memIndex = bx*TILE_WIDTH + tx;
if (memIndex < len)							//out of bounds checking - some threads will be doing nothing
{
result = 0;
dAbs = 5;
sign = 1;

while (result != -1)				//if result != -1, increment d and try again
{
n1 = nArray[memIndex];				//reinitialize n1 to n
d = dAbs*sign;
t = 1;
d1 = d;							//reinitialize d1 to d
d1 = d1 % n1;

while (d1 != 0)
{
while (d1 % 2 == 0)        //while d is even
{
d1 = d1 / 2;
if (n1 % 8 == 3 || n1 % 8 == 5) t = -t;
}
temp = d1;
d1 = n1;
n1 = temp;
if ((d1 % 4 == 3) && (n1 % 4 == 3)) t = -t;
d1 = d1 % n1;
}
if (n1 == 1) result = t;
else result = 0;
dAbs = dAbs + 2;
sign = sign * -1;
}
}
__syncthreads();
if (memIndex < len)
dArray[memIndex] = d;
__syncthreads();
}