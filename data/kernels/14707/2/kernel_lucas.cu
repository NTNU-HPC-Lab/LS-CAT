#include "hip/hip_runtime.h"
#include "includes.h"
/*****************************************************************************/
// nvcc -O1 -o bpsw bpsw.cu -lrt -lm


// Assertion to check for errors
__global__ void kernel_lucas(long* nArray, long* dArray, int* rArray, long len) {
int bx = blockIdx.x;      // ID thread
int tx = threadIdx.x;
int i, length;
long long d, n;
long long q, q2, u, u2, uold, v, v2, t;

// Identify the row and column of the Pd element to work on
long memIndex = bx*TILE_WIDTH + tx;
if (memIndex < len)							//out of bounds checking - some threads will be doing nothing
{
d = (long long) dArray[memIndex];
n = (long long) nArray[memIndex];
q = (1 - d) / 4;
u = 0;
v = 2;
u2 = 1;
v2 = 1;
q2 = 2 * q;
t = (n + 1) / 2;						//theta
length = 32 - __clz(t); //length of our number in bits. //clz(b00010010) = 3

for (i = 0; i < length; i++)
{
u2 = (u2 * v2) % n;
v2 = (v2 * v2 - q2) % n;
if (t & 1)				//mask = 1
{
uold = u;
u = (u2 * v) + (u * v2);
u = (u % 2 == 1) ? u + n : u;
u = (u / 2) % n;
v = (v2 * v) + (u2 * uold * d);
v = (v % 2 == 1) ? v + n : v;
v = (v / 2) % n;
}

q = (q*q) % n;
q2 = q + q;

t = t >> 1;
}

}
__syncthreads();
if (memIndex < len)
rArray[memIndex] = (u == 0);

}