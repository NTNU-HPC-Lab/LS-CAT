#include "hip/hip_runtime.h"
#include "includes.h"

/// Tile size used by the OptimizedMMKernel
#define TILE_SIZE 32

/// Naive matrix multiplication CUDA Kernel

/// Tiled 1D Shared Memory No Unrolling

/// Tiled 2D Shared Memory No Unrolling

/// Tiled 2D Shared Memory With Unrolling (4x4 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (8x8 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (16x16 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (32x32 Tile Size)

/// Prints a matrix out to the stderr stream
__global__ void OptimizedMMKernel_2_32(float *a, float *b, float *c, int size)
{
// Create shared matrices for rows of A and columns of B
__shared__ float sharedA[32][32];
__shared__ float sharedB[32][32];

int tx = threadIdx.x;
int ty = threadIdx.y;

int x = blockIdx.x * blockDim.x + tx;
int y = blockIdx.y * blockDim.y + ty;

float sum = 0;

// Divide the matrix up into tiles based on the tile size so each thread
// Can perform its partial sum of the dot product from the shared matrix
int tilesPerGrid = size / blockDim.x;
for (int i = 0; i < tilesPerGrid; i++)
{
// Each thread loads element into A and B
sharedA[ty][tx] = a[(y * size) + (i * 32) + tx];
sharedB[ty][tx] = b[(i * 32 * size) + (ty * size) + x];

// Wait for all threads to load each section of the shared matrix
__syncthreads();

sum += sharedA[ty][0] * sharedB[0][tx];
sum += sharedA[ty][1] * sharedB[1][tx];
sum += sharedA[ty][2] * sharedB[2][tx];
sum += sharedA[ty][3] * sharedB[3][tx];
sum += sharedA[ty][4] * sharedB[4][tx];
sum += sharedA[ty][5] * sharedB[5][tx];
sum += sharedA[ty][6] * sharedB[6][tx];
sum += sharedA[ty][7] * sharedB[7][tx];
sum += sharedA[ty][8] * sharedB[8][tx];
sum += sharedA[ty][9] * sharedB[9][tx];
sum += sharedA[ty][10] * sharedB[10][tx];
sum += sharedA[ty][11] * sharedB[11][tx];
sum += sharedA[ty][12] * sharedB[12][tx];
sum += sharedA[ty][13] * sharedB[13][tx];
sum += sharedA[ty][14] * sharedB[14][tx];
sum += sharedA[ty][15] * sharedB[15][tx];
sum += sharedA[ty][16] * sharedB[16][tx];
sum += sharedA[ty][17] * sharedB[17][tx];
sum += sharedA[ty][18] * sharedB[18][tx];
sum += sharedA[ty][19] * sharedB[19][tx];
sum += sharedA[ty][20] * sharedB[20][tx];
sum += sharedA[ty][21] * sharedB[21][tx];
sum += sharedA[ty][22] * sharedB[22][tx];
sum += sharedA[ty][23] * sharedB[23][tx];
sum += sharedA[ty][24] * sharedB[24][tx];
sum += sharedA[ty][25] * sharedB[25][tx];
sum += sharedA[ty][26] * sharedB[26][tx];
sum += sharedA[ty][27] * sharedB[27][tx];
sum += sharedA[ty][28] * sharedB[28][tx];
sum += sharedA[ty][29] * sharedB[29][tx];
sum += sharedA[ty][30] * sharedB[30][tx];
sum += sharedA[ty][31] * sharedB[31][tx];

// Wait for all threads to compute their partial sum from the shared matrices before loading the next
__syncthreads();
}

// Store the full sum as the result
c[y * size + x] = sum;
}