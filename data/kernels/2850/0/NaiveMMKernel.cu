#include "hip/hip_runtime.h"
#include "includes.h"

/// Tile size used by the OptimizedMMKernel
#define TILE_SIZE 32

/// Naive matrix multiplication CUDA Kernel

/// Tiled 1D Shared Memory No Unrolling

/// Tiled 2D Shared Memory No Unrolling

/// Tiled 2D Shared Memory With Unrolling (4x4 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (8x8 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (16x16 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (32x32 Tile Size)

/// Prints a matrix out to the stderr stream
__global__ void NaiveMMKernel(float *a, float *b, float *c, int size)
{
int xOut = blockDim.x * blockIdx.x + threadIdx.x;
int yOut = blockDim.y * blockIdx.y + threadIdx.y;

float outValue = 0;
for (int i = 0; i < size; i++)
{
// Row of a mulitplied by the column of b
float prod = a[yOut * size + i] * b[i * size + xOut];
outValue += prod;
}

// Store sum of dot products in C matrix
c[yOut * size + xOut] = outValue;
}