#include "hip/hip_runtime.h"
#include "includes.h"

/// Tile size used by the OptimizedMMKernel
#define TILE_SIZE 32

/// Naive matrix multiplication CUDA Kernel

/// Tiled 1D Shared Memory No Unrolling

/// Tiled 2D Shared Memory No Unrolling

/// Tiled 2D Shared Memory With Unrolling (4x4 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (8x8 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (16x16 Tile Size)

/// Tiled 2D Shared Memory With Unrolling (32x32 Tile Size)

/// Prints a matrix out to the stderr stream
__global__ void OptimizedMMKernel_1(float *a, float *b, float *c, int size)
{
// Create shared matrices for rows of A and columns of B
__shared__ float sharedA[TILE_SIZE][TILE_SIZE];
__shared__ float sharedB[TILE_SIZE][TILE_SIZE];

int tx = threadIdx.x;
int ty = threadIdx.y;

int x = blockIdx.x * blockDim.x + tx;
int y = blockIdx.y * blockDim.y + ty;

float sum = 0;

// Divide the matrix up into tiles based on the tile size so each thread
// Can perform its partial sum of the dot product from the shared matrix
int tilesPerGrid = size / blockDim.x;
for (int i = 0; i < tilesPerGrid; i++)
{
// Each thread loads element into A and B
sharedA[ty][tx] = a[(y * size) + (i * TILE_SIZE) + tx];
sharedB[ty][tx] = b[(i * TILE_SIZE * size) + (ty * size) + x];

// Wait for all threads to load each section of the shared matrix
__syncthreads();

for (int j = 0; j < TILE_SIZE; j++)
{
sum += sharedA[ty][j] * sharedB[j][tx];
}

// Wait for all threads to compute their partial sum from the shared matrices before loading the next
__syncthreads();
}

// Store the full sum as the result
c[y * size + x] = sum;
}