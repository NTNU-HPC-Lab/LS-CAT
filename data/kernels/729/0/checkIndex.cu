#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex(void){
printf("threadIdx: (%d,%d,%d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n",
threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z,
blockDim.x, blockDim.y, blockDim.z,
gridDim.x, gridDim.y,gridDim.z
);
}