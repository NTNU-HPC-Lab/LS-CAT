#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrayOnGPU(float *A, float *B, float *C){
// スレッドIDを割り当てる
int i = threadIdx.x;
C[i] = A[i] + B[i];
}