#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
int Row = blockIdx.y*blockDim.y+threadIdx.y;// Calculate the row index of the P element and M
int Col = blockIdx.x*blockDim.x+threadIdx.x;// Calculate the column index of P and N
if ((Row < Width) && (Col < Width)) {
float Pvalue = 0;
for (int k = 0; k < Width; ++k) {
Pvalue += M[Row*Width+k]*N[k*Width+Col];// each thread computes one element of the block sub-matrix
}

P[Row*Width+Col] = Pvalue;
}
}