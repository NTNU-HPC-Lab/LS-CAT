#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
C[i] = A[i] + B[i];
}
/* printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
"gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
gridDim.x,gridDim.y,gridDim.z);*/
}