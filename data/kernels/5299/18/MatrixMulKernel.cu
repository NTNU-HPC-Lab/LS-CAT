#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
//We need to iterate with tiles - starting point and end needed for tiles
int Mstart=Width*BLOCK_SIZE*blockIdx.y;//rows of matrix M
int Mend=Mstart+Width-1;
int mstep=BLOCK_SIZE;
int Nstart=BLOCK_SIZE*blockIdx.x;//cols of matrix N
int nstep=BLOCK_SIZE*Width;
float temp=0;

//loop through tiles


for(int m=Mstart,n=Nstart;m<Mend;m+=mstep,n+=nstep){
__shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];
Ms[threadIdx.y][threadIdx.x]=M[m+Width*threadIdx.y+threadIdx.x];
Ns[threadIdx.y][threadIdx.x]=N[n+Width*threadIdx.y+threadIdx.x];
__syncthreads();


for (int i = 0; i < BLOCK_SIZE; ++i) {
temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
}

__syncthreads();

}

P[Width * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x + Width * threadIdx.y + threadIdx.x] = temp;

}