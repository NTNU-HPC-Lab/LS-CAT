#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const double *A, const double *B, double *C, int numElements)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < numElements)
{
C[i] = A[i] + B[i];
}
printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
"gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
gridDim.x,gridDim.y,gridDim.z);
}