#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;



__global__ void matrixMultiplicationKernel(long* A, long* B, long* C, long N) {

long ROW = (blockIdx.y*blockDim.y) + threadIdx.y;
long COL = (blockIdx.x*blockDim.x) + threadIdx.x;

long tmpSum = 0;

if (ROW < N && COL < N) {
// each thread computes one element of the block sub-matrix
for (long i = 0; i < N; i++) {
tmpSum += A[ROW * N + i] * B[i * N + COL];
}

C[ROW * N + COL] = tmpSum;
}
}