#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void addKernel(int *c, const int *a, const int *b, int size) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < size) {
c[i] = a[i] + b[i];
}
}