#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xcorr(float *d_i1, float *d_i2, float *d_icorr, int m1, int n1, int m2, int n2)
{
//grab theadID location
int tx = threadIdx.x;
int ty = threadIdx.y;

// get output values based on block and thread locations
int row_out = blockIdx.y*blockDim.y + ty;
int col_out = blockIdx.x*blockDim.x + tx;

// Get starting value for the convolution as dictated by m2 and n2
// we'll use i1 indicies as the coord syst.
int row_st = row_out - (m2 - 1);
int col_st = col_out - (n2 - 1);

// correlation variable
float corr=0; // initialize correlation variable

if (row_out >= 0 && row_out < m1 + m2 - 1 && col_out >= 0 && col_out < n1 + n2 - 1)  // ensure output is within bounds of correlation image
{
// Buffering into memory would be 1 call to a global variable, From there we need 1 call for each multiplication, however we only need to make 1 call to a global
// variable for the multiplication and move on, as such it doesn't make sense to buffer these images into local memory
for (int i = 0; i < m2; i++) { //
for (int j = 0; j < n2; j++)
{
if (row_st + i >= 0 && row_st + i < m1 && col_st + j >= 0 && col_st + j < n1) { // if row start and col start are greater than 0 and less than the number of pixels available perform convolution
corr += d_i1[row_st + i + (col_st + j) * m1] * d_i2[i + j * m2];
}
else {} // if else is invoked it's because row_st and col_st are outside of im1 bounds and the convolution should be left alone
}
}
d_icorr[row_out + col_out*(m1 + m2 - 1)] = corr; // assign correlation variable to proper location in final image
}
else{}

}