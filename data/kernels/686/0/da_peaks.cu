#include "hip/hip_runtime.h"
#include "includes.h"
/*
* get_da_peaks is a gpu_accelerated local maxima finder
* [iprod] = get_da_peaks(i1, r, thresh);
* Written by Andrew Nelson 7/20/17
*
*
*
*
*/

// includes, project


// main
__global__ void da_peaks(float *d_i1, float thresh, int m, int n, int o)
{

int tx = threadIdx.x;
int ty = threadIdx.y;
float d_i2[25];
// location of output pixel being analyzed
int row_output = blockIdx.y*blockDim.y + ty;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
int col_output = blockIdx.x*blockDim.x + tx;		// gives x coordinate as a function of tile width
int imnum = blockIdx.z;
if (imnum < o && row_output >=2 && row_output < m-2 && col_output >=2 && col_output <n-2)
{
// buffer the info into
for(int i = 0; i <5 ; i++){
for(int j = 0; j <5 ; j++)
{
d_i2[i*5 + j] = d_i1[(row_output - 2 + i) + (col_output - 2 +j)*m + imnum*m*n];
}
}
float me = d_i2[12];
int maxi = 1;
if(me < thresh){maxi = 0;}
for(int k = 0; k <25; k++)
{
if(d_i2[k] > me){maxi = 0;}
}
d_i1[row_output + col_output*m + imnum*m*n] = maxi;
}
else if(imnum <o){d_i1[row_output + col_output*m + imnum*m*n] = 0;}
else{}
}