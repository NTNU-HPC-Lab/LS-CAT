#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void drift(float *d_i1, float *d_icorr, int m1, int n1, int o1)
{
//grab theadID location
int tx = threadIdx.x;
int ty = threadIdx.y;

// get output values based on block and thread locations
int row_out = blockIdx.y*blockDim.y + ty;
int col_out = blockIdx.x*blockDim.x + tx;
int im_out  = blockIdx.z;

// Get starting value for the convolution as dictated by m2 and n2
// we'll use i1 indicies as the coord syst.
int row_st = row_out - (m1 - 1);
int col_st = col_out - (n1 - 1);

// correlation variable
float corr=0; // initialize correlation variable

if (row_out >= 0 && row_out < 2*m1 - 1 && col_out >= 0 && col_out < 2*n1- 1 && im_out < o1-1)  // ensure output is within bounds of correlation image
{
// Buffering into memory would be 1 call to a global variable, From there we need 1 call for each multiplication, however we only need to make 1 call to a global
// variable for the multiplication and move on, as such it doesn't make sense to buffer these images into local memory
for (int i = 0; i < m1; i++) { //
for (int j = 0; j < n1; j++)
{
if (row_st + i >= 0 && row_st + i < m1 && col_st + j >= 0 && col_st + j < n1) { // if row start and col start are greater than 0 and less than the number of pixels available perform convolution
corr += d_i1[row_st + i + (col_st + j) * m1 + im_out*m1*n1] * d_i1[i + j * m1 + (im_out+1)*m1*n1]; // shift n+1 image over n image
}
else {} // if else is invoked it's because row_st and col_st are outside of im1 bounds and the convolution should be left alone
}
}
d_icorr[row_out + col_out*(2*m1 - 1) + im_out*(2*m1 - 1)*(2*n1 - 1)] = corr; // assign correlation variable to proper location in final image
}
else{}


}