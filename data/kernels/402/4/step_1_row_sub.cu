#include "hip/hip_runtime.h"
#include "includes.h"
// Fast Block Distributed CUDA Implementation of the Hungarian Algorithm
//
// Annex to the paper:
// Paulo A. C. Lopes, Satyendra Singh Yadav, Aleksandar Ilic, Sarat Kumar Patra ,
// "Fast Block Distributed CUDA Implementation of the Hungarian Algorithm",
// Journal Parallel Distributed Computing
//
// Hungarian algorithm:
// (This algorithm was modified to result in an efficient GPU implementation, see paper)
//
// Initialize the slack matrix with the cost matrix, and then work with the slack matrix.
//
// STEP 1: Subtract the row minimum from each row. Subtract the column minimum from each column.
//
// STEP 2: Find a zero of the slack matrix. If there are no starred zeros in its column or row star the zero.
// Repeat for each zero.
//
// STEP 3: Cover each column with a starred zero. If all the columns are
// covered then the matching is maximum.
//
// STEP 4: Find a non-covered zero and prime it. If there is no starred zero in the row containing this primed zero,
// Go to Step 5. Otherwise, cover this row and uncover the column containing the starred zero.
// Continue in this manner until there are no uncovered zeros left.
// Save the smallest uncovered value and Go to Step 6.
//
// STEP 5: Construct a series of alternating primed and starred zeros as follows:
// Let Z0 represent the uncovered primed zero found in Step 4.
// Let Z1 denote the starred zero in the column of Z0(if any).
// Let Z2 denote the primed zero in the row of Z1(there will always be one).
// Continue until the series terminates at a primed zero that has no starred zero in its column.
// Un-star each starred zero of the series, star each primed zero of the series,
// erase all primes and uncover every row in the matrix. Return to Step 3.
//
// STEP 6: Add the minimum uncovered value to every element of each covered row,
// and subtract it from every element of each uncovered column.
// Return to Step 4 without altering any stars, primes, or covered rows.


// Uncomment to use chars as the data type, otherwise use int
// #define CHAR_DATA_TYPE

// Uncomment to use a 4x4 predefined matrix for testing
// #define USE_TEST_MATRIX

// Comment to use managed variables instead of dynamic parallelism; usefull for debugging
// #define DYNAMIC

#define klog2(n) ((n<8)?2:((n<16)?3:((n<32)?4:((n<64)?5:((n<128)?6:((n<256)?7:((n<512)?8:((n<1024)?9:((n<2048)?10:((n<4096)?11:((n<8192)?12:((n<16384)?13:0))))))))))))

#ifndef DYNAMIC
#define MANAGED __managed__
#define dh_checkCuda checkCuda
#define dh_get_globaltime get_globaltime
#define dh_get_timer_period get_timer_period
#else
#define dh_checkCuda d_checkCuda
#define dh_get_globaltime d_get_globaltime
#define dh_get_timer_period d_get_timer_period
#define MANAGED
#endif

#define kmin(x,y) ((x<y)?x:y)
#define kmax(x,y) ((x>y)?x:y)

#ifndef USE_TEST_MATRIX
#ifdef _n_
// These values are meant to be changed by scripts
const int n = _n_;							// size of the cost/pay matrix
const int range = _range_;					// defines the range of the random matrix.
const int user_n = n;
const int n_tests = 100;
#else
// User inputs: These values should be changed by the user
const int user_n = 1000;				// This is the size of the cost matrix as supplied by the user
const int n = 1<<(klog2(user_n)+1);		// The size of the cost/pay matrix used in the algorithm that is increased to a power of two
const int range = n;					// defines the range of the random matrix.
const int n_tests = 10;					// defines the number of tests performed
#endif

// End of user inputs

const int log2_n = klog2(n);			// log2(n)
const int n_threads = kmin(n,64);		// Number of threads used in small kernels grid size (typically grid size equal to n)
// Used in steps 3ini, 3, 4ini, 4a, 4b, 5a and 5b (64)
const int n_threads_reduction = kmin(n, 256);		// Number of threads used in the redution kernels in step 1 and 6 (256)
const int n_blocks_reduction = kmin(n, 256);		// Number of blocks used in the redution kernels in step 1 and 6 (256)
const int n_threads_full = kmin(n, 512);			// Number of threads used the largest grids sizes (typically grid size equal to n*n)
// Used in steps 2 and 6 (512)
const int seed = 45345;					// Initialization for the random number generator

#else
const int n = 4;
const int log2_n = 2;
const int n_threads = 2;
const int n_threads_reduction = 2;
const int n_blocks_reduction = 2;
const int n_threads_full = 2;
#endif

const int n_blocks = n / n_threads;									// Number of blocks used in small kernels grid size (typically grid size equal to n)
const int n_blocks_full = n * n / n_threads_full;					// Number of blocks used the largest gris sizes (typically grid size equal to n*n)
const int row_mask = (1 << log2_n) - 1;								// Used to extract the row from tha matrix position index (matrices are column wise)
const int nrows = n, ncols = n;										// The matrix is square so the number of rows and columns is equal to n
const int max_threads_per_block = 1024;								// The maximum number of threads per block
const int columns_per_block_step_4 = 512;							// Number of columns per block in step 4
const int n_blocks_step_4 = kmax(n / columns_per_block_step_4, 1);	// Number of blocks in step 4 and 2
const int data_block_size = columns_per_block_step_4 * n;			// The size of a data block. Note that this can be bigger than the matrix size.
const int log2_data_block_size = log2_n + klog2(columns_per_block_step_4);	// log2 of the size of a data block. Note that klog2 cannot handle very large sizes

// For the selection of the data type used
#ifndef CHAR_DATA_TYPE
typedef int data;
#define MAX_DATA INT_MAX
#define MIN_DATA INT_MIN
#else
typedef unsigned char data;
#define MAX_DATA 255
#define MIN_DATA 0
#endif

// Host Variables

// Some host variables start with h_ to distinguish them from the corresponding device variables
// Device variables have no prefix.

#ifndef USE_TEST_MATRIX
data h_cost[ncols][nrows];
#else
data h_cost[n][n] = { { 1, 2, 3, 4 }, { 2, 4, 6, 8 }, { 3, 6, 9, 12 }, { 4, 8, 12, 16 } };
#endif
int h_column_of_star_at_row[nrows];
int h_zeros_vector_size;
int h_n_matches;
bool h_found;
bool h_goto_5;

// Device Variables

__device__ data slack[nrows*ncols];						// The slack matrix
__device__ data min_in_rows[nrows];						// Minimum in rows
__device__ data min_in_cols[ncols];						// Minimum in columns
__device__ int zeros[nrows*ncols];						// A vector with the position of the zeros in the slack matrix
__device__ int zeros_size_b[n_blocks_step_4];			// The number of zeros in block i

__device__ int row_of_star_at_column[ncols];			// A vector that given the column j gives the row of the star at that column (or -1, no star)
__device__ int column_of_star_at_row[nrows];			// A vector that given the row i gives the column of the star at that row (or -1, no star)
__device__ int cover_row[nrows];						// A vector that given the row i indicates if it is covered (1- covered, 0- uncovered)
__device__ int cover_column[ncols];						// A vector that given the column j indicates if it is covered (1- covered, 0- uncovered)
__device__ int column_of_prime_at_row[nrows];			// A vector that given the row i gives the column of the prime at that row  (or -1, no prime)
__device__ int row_of_green_at_column[ncols];			// A vector that given the row j gives the column of the green at that row (or -1, no green)

__device__ data max_in_mat_row[nrows];					// Used in step 1 to stores the maximum in rows
__device__ data min_in_mat_col[ncols];					// Used in step 1 to stores the minimums in columns
__device__ data d_min_in_mat_vect[n_blocks_reduction];	// Used in step 6 to stores the intermediate results from the first reduction kernel
__device__ data d_min_in_mat;							// Used in step 6 to store the minimum

MANAGED __device__ int zeros_size;					// The number fo zeros
MANAGED __device__ int n_matches;					// Used in step 3 to count the number of matches found
MANAGED __device__ bool goto_5;						// After step 4, goto step 5?
MANAGED __device__ bool repeat_kernel;				// Needs to repeat the step 2 and step 4 kernel?
#if defined(DEBUG) || defined(_DEBUG)
MANAGED __device__ int n_covered_rows;				// Used in debug mode to check for the number of covered rows
MANAGED __device__ int n_covered_columns;			// Used in debug mode to check for the number of covered columns
#endif

__shared__ extern data sdata[];							// For access to shared memory

// -------------------------------------------------------------------------------------
// Device code
// -------------------------------------------------------------------------------------

__global__ void step_1_row_sub()
{

int i = blockDim.x * blockIdx.x + threadIdx.x;
int l = i & row_mask;

slack[i] = slack[i] - min_in_rows[l];  // subtract the minimum in row from that row

}