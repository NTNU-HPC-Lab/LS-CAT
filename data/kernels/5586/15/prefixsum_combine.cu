#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixsum_combine(float* in, int in_length, float* out, int out_length){

int idx = blockDim.x * blockIdx.x + threadIdx.x;

if(idx < out_length && blockIdx.x > 0){
out[idx] += in[blockIdx.x - 1];
}

}