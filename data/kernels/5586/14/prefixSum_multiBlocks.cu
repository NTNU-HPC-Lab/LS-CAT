#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSum_multiBlocks(float* in, int in_length, float* out, float* temp ){

extern __shared__ float DSM[];

int idx = blockIdx.x * blockDim.x + threadIdx.x;

//load in shared memory
if(idx < in_length){
DSM[threadIdx.x] = in[idx];

//partial sums phase
for(int stride = 1; stride <= blockDim.x; stride *= 2){
__syncthreads();
int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
if(index_aux < blockDim.x)
DSM[index_aux] += DSM[index_aux - stride];
}

//reduction phase
for(int stride=blockDim.x/4 ; stride > 0 ; stride /= 2){
__syncthreads();

int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
if(index_aux + stride < blockDim.x)
DSM[index_aux + stride] += DSM[index_aux];
}

__syncthreads();

//save intermediary values on temp to post combine for multi blocks
if(threadIdx.x == 0)
temp[blockIdx.x] = DSM[blockDim.x - 1];

out[idx] = DSM[threadIdx.x];

}

}