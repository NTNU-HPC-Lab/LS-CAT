#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulKernel(float* d_M, float* d_N, float* d_P, int width){

//compute row and column of the target element to compute
int row = blockDim.y * blockIdx.y + threadIdx.y;
int col = blockDim.x * blockIdx.x + threadIdx.x;

//check for safety if target element is within matrix dimensions
if(row < width && col < width){
//perform "dot product" line and column
float sum = 0.0f;
for (int k = 0; k < width; ++k) {
sum += d_M[row * width + k] * d_N[k * width + col];
}
//assign target element value
d_P[row * width + col] = sum;
}
}