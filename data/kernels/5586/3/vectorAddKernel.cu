#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddKernel(float* inputA, float* inputB, float* output, int length){

//compute element index
int idx = blockIdx.x * blockDim.x + threadIdx.x;

//add an vector element
if(idx < length) output[idx] = inputA[idx] + inputB[idx];

}