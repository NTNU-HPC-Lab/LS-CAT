#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reductionKernel(float* vec, int width, double* sumUp){

//shared memory instantiation
extern __shared__ float partialSum[];

//index for global memory
int g_idx = blockDim.x * blockIdx.x + threadIdx.x;
//index for shared memory
int b_idx = threadIdx.x;

//load shared memory from global memory
partialSum[b_idx] = g_idx < width ? vec[g_idx] : 0;

//reduction inside blocks
for(int stride = blockDim.x/2; stride >= 1 ; stride = stride/2){

__syncthreads();
if(b_idx < stride ){
partialSum[b_idx] = partialSum[b_idx] + partialSum[b_idx + stride];
}
}

//reduction for grid using just thread 0 of each block
if(b_idx == 0){
//coppy value back to global memory
vec[g_idx] = partialSum[b_idx];

//reduction
for(int stride = (gridDim.x * blockDim.x)/2; stride>=blockDim.x; stride = stride/2){

__syncthreads();
if(g_idx < stride){
vec[g_idx] = vec[g_idx] + vec[g_idx + stride];
}
}
}

//save result in output variable
if(g_idx == 0)
(*sumUp) = vec[g_idx];
}