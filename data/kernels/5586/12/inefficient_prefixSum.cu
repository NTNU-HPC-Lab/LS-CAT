#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inefficient_prefixSum(float* in, int in_length, float* out ){

//shared memory declaration
extern __shared__ float DSM[];

//compute index
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if(idx < in_length){
//load on shared memory
DSM[threadIdx.x] = in[idx];

//compute prefix_sum making sequence of sums
for(int stride = 1; stride <= threadIdx.x; stride *= 2){
__syncthreads();

DSM[threadIdx.x] =  DSM[threadIdx.x] + DSM[threadIdx.x - stride];
}

out[idx] = DSM[threadIdx.x];

}

}