#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd_B_Kernel(float* A, float* B, float* C, size_t pitch, int width){

//compute indexes
int row = blockIdx.x * blockDim.x + threadIdx.x;


int rowWidthWithPad = pitch/sizeof(float);

if(row < width){
for (int col = 0; col < width; ++col) {
if(col < width)
C[row * rowWidthWithPad  + col] = A[row * rowWidthWithPad  + col] + B[row * rowWidthWithPad  + col];
}
}


}