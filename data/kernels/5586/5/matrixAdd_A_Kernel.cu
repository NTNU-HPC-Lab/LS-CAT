#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd_A_Kernel(float* A, float* B, float* C, size_t pitch, int width){

//compute indexes
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

int rowWidthWithPad = pitch/sizeof(float);


if(row < width && col < width)
C[row * rowWidthWithPad  + col] = A[row * rowWidthWithPad  + col] + B[row * rowWidthWithPad  + col];

}