#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixVectorMultKernel(float* fltMatrix, float* vec, float* output, int rows, int columns){

int row = blockDim.x * blockIdx.x + threadIdx.x;

if(row < rows){
float sum = 0.0f;
for (int col = 0; col < columns; ++col) {
sum += fltMatrix[row * columns + col] + vec[col];
}

output[row] = sum;
}

}