#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd_C_Kernel(float* A, float* B, float* C, size_t pitch, int width){

//compute indexes
int col = blockIdx.x * blockDim.x + threadIdx.x;


int rowWidthWithPad = pitch/sizeof(float);

if(col < width){
for (int row = 0; row < width; ++row) {
if(row < width)
C[row * rowWidthWithPad  + col] = A[row * rowWidthWithPad  + col] + B[row * rowWidthWithPad  + col];
}
}


}