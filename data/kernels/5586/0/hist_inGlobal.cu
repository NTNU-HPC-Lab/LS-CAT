#include "hip/hip_runtime.h"
#include "includes.h"

//kernel for computing histogram right in memory

//computer partial histogram on shared memory and mix them on global memory

__global__ void hist_inGlobal (const int* values, int length, int* hist){

//compute index and interval
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;
//iterate over index and interval since it is less than the total length
while(idx < length){
//get value
int val = values[idx];
//increment value frequency on histogram using atomic in order to be thread safe
atomicAdd(&hist[val], 1);
idx += stride;
}
}