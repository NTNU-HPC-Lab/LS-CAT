#include "hip/hip_runtime.h"
#include "includes.h"

//kernel for computing histogram right in memory

//computer partial histogram on shared memory and mix them on global memory

__global__ void hist_inShared (const int* values, int length, int* hist){

//load shared memory
extern __shared__ int shHist[];
shHist[threadIdx.x] = 0;
__syncthreads();

//compute index and interval
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;

//iterate over index and interval since it is less than the total length
while(idx < length){
int val = values[idx];
//increment value frequency on histogram using atomic in order to be thread safe
atomicAdd(&shHist[val], 1);
idx += stride;
}

//combine partial histogram on shared memory to create a full histogram
__syncthreads();
atomicAdd(&hist[threadIdx.x], shHist[threadIdx.x]);
}