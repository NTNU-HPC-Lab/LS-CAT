#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSum_UniqueBlock(float* in, int in_length, float* out ){

//shared memory declaration
extern __shared__ float DSM[];

int idx = blockIdx.x * blockDim.x + threadIdx.x;

//load in shared memory
if(idx < in_length){
DSM[threadIdx.x] = in[idx];

//partial sums phase
for(int stride = 1; stride <= blockDim.x; stride *= 2){
__syncthreads();
int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
if(index_aux < blockDim.x)
DSM[index_aux] += DSM[index_aux - stride];
}

//reduction phase
for(int stride=blockDim.x/4 ; stride > 0 ; stride /= 2){
__syncthreads();

int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
if(index_aux + stride < blockDim.x)
DSM[index_aux + stride] += DSM[index_aux];
}

__syncthreads();

out[idx] = DSM[threadIdx.x];

}

}