#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_down(float *d_out, float *d_in, const int delta) {
float value = d_in[threadIdx.x];
value = __shfl_down_sync(0xffffffff,value,delta,16);
d_out[threadIdx.x] = value;
}