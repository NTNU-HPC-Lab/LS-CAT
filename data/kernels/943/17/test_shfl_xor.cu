#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_xor(float *d_out, float *d_in, const int mask) {
float value = d_in[threadIdx.x];
value = __shfl_xor_sync(0xffffffff,value,mask,BDIMX);
d_out[threadIdx.x] = value;
}