#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(float *loc, float *temp, const int num) {
int idx = blockIdx.x*blockDim.x+threadIdx.x;
if(idx < num) {
atomicAdd(loc,temp[idx]);
}
}