#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_broadcast(float *d_out, float *d_in, const int srcLane) {
float value = d_in[threadIdx.x];
value = __shfl_sync(0xffffffff,value,srcLane,BDIMX);
d_out[threadIdx.x] = value;
}