#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_shfl_up(float *d_out, float *d_in, const int delta) {
float value = d_in[threadIdx.x];
value = __shfl_up_sync(0xffffffff,value,delta,16);
d_out[threadIdx.x] = value;
}