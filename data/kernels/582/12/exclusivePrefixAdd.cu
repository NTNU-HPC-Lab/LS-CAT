#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exclusivePrefixAdd(unsigned int* d_in, unsigned int* d_out)
{
//Belloch implementation
//NOTE: this is set up specifically for 1 block of 1024 threads

int thread_x = threadIdx.x;

d_out[thread_x] = d_in[thread_x];
__syncthreads();

//first, do the reduce:
for (unsigned int i = 2; i <= blockDim.x; i <<= 1)
{
if ((thread_x + 1) % i == 0)
{
d_out[thread_x] = d_out[thread_x] + d_out[thread_x - i / 2];
}

__syncthreads();
}


//now do the downsweep part:

if (thread_x == blockDim.x - 1)
{
d_out[thread_x] = 0;
}

//maybe need a syncthreads() here because of that write above? it's only 1 thread so idk if it affects it

for (unsigned int i = blockDim.x; i >= 2; i >>= 1)
{
if ((thread_x + 1) % i == 0)
{
unsigned int temp = d_out[thread_x - (i / 2)];

//the "left" copy
d_out[thread_x - (i / 2)] = d_out[thread_x];

//and the "right" operation
d_out[thread_x] = temp + d_out[thread_x];
}
__syncthreads();
}

}