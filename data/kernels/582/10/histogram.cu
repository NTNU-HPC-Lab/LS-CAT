#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram(const float* d_in, unsigned int* d_out, const float lumMin, const float lumRange, const size_t numBins, const size_t size)
{
int abs_x = threadIdx.x + blockDim.x * blockIdx.x;

if (abs_x > size)
{
return;
}

int bin = (d_in[abs_x] - lumMin) / lumRange * numBins;

//then increment:
atomicAdd(&(d_out[bin]), 1);
}