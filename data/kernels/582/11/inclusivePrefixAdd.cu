#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inclusivePrefixAdd(unsigned int* d_in, unsigned int* d_out)
{
//Hillis Steele implementation
//NOTE: right now, this is only set up for 1 block of 1024 threads

int abs_x = threadIdx.x + blockIdx.x * blockDim.x;
int thread_x = threadIdx.x;

extern __shared__ unsigned int segment[];
segment[thread_x] = d_in[abs_x];
//d_out[thread_x] = d_in[thread_x];
__syncthreads();

for (unsigned int i = 1; i < blockDim.x; i <<= 1)
{
if (thread_x >= i)
{
//d_out[thread_x] = d_out[thread_x] + d_out[thread_x - i];
segment[thread_x] = segment[thread_x] + segment[thread_x - i];
}

__syncthreads();
}

//this happens in different blocks, so no need to syncthreads()
if (blockIdx.x > 0)
{
//carry over the result of the last segment
segment[thread_x] = segment[thread_x] + d_out[blockDim.x * (blockIdx.x - 1)];
}

d_out[abs_x] = segment[thread_x];
}