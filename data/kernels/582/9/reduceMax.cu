#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceMax(const float* d_in, float* d_out)
{
int abs_x = threadIdx.x + blockIdx.x * blockDim.x;
int thread_x = threadIdx.x;

extern __shared__ float sdata[];

sdata[thread_x] = d_in[abs_x];
__syncthreads();

int last_i = blockDim.x;
for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1)
{
if (thread_x < i)
{
sdata[thread_x] = max(sdata[thread_x], sdata[thread_x + i]);

//this checks for weird edge cases where the block dimension is not a power of two
//see https://discussions.udacity.com/t/wrong-max-value-problem-set-3/85232/7

//basically, if we are at the "last" thread of this iteration (i - 1)
//and if we lost a point due to integer divison
if (thread_x == i - 1 && last_i > 2 * i)
{
//then take the point we lost to integer divison at (last_i - 1)
sdata[thread_x] = max(sdata[thread_x], sdata[last_i - 1]);
}
}

__syncthreads();
last_i = i;
}

//return result at the 0th thread of every block:
if (thread_x == 0)
{
d_out[blockIdx.x] = sdata[0];
}
}