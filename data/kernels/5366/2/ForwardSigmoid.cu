#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardSigmoid(float* Z, int nRowsZ, int nColsZ, float* A)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < nRowsZ * nColsZ)
{
A[index] = 1 / (1 + exp(-Z[index]));
}
}