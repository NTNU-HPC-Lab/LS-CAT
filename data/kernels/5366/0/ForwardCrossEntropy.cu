#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardCrossEntropy(float *output, float *labels, int nColsOutput, float *loss)
{
int col = blockIdx.x;

float temp = -(labels[col] * logf(output[col]) + logf(1 - output[col])
* (1 - labels[col]));
atomicAdd(loss, temp);
}