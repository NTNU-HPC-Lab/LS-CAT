#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BackwardSoftmax(float *A, float *dA, int nColsdZ, float *dZ)
{
int row = threadIdx.x;
int col = blockIdx.x;

dZ[row * nColsdZ + col] = dA[row * nColsdZ + col] * A[row * nColsdZ + col] *
(1 - A[row * nColsdZ + col]);
}