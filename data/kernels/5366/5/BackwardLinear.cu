#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BackwardLinear(float *dZ, float *W, int nColsW, int nRowsW, int nColsdZ, float *dA)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

float dAValue = 0;

if (row < nColsW && col < nColsdZ)
{
for (int i = 0; i < nRowsW; i++)
{
dAValue += W[i * nColsW + row] * dZ[i * nColsdZ + col];
}
dA[row * nColsdZ + col] = dAValue;
}
}