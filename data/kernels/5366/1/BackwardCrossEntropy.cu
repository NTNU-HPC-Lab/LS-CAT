#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BackwardCrossEntropy(float *output, float *labels, int nColsOutput, float *dOutput)
{
int col = blockIdx.x;

dOutput[col] = (labels[col] / output[col] - (1 - labels[col]) /
(1 - output[col])) * -1;
}