#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BackwardSigmoid(float* Z, float* dA, int nRowsdZ, int nColsdZ, float *dZ)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < nRowsdZ * nColsdZ)
{
dZ[index] = 1 / (1 + exp(-Z[index])) * (1 - 1 / (1 + exp(-Z[index]))) *
dA[index];
}
}