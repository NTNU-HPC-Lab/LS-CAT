#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BackwardReLU(float* Z, float* dA, int nRowsdZ, int nColsdZ, float *dZ)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < nRowsdZ * nColsdZ)
{
if (Z[index] >= 0)
dZ[index] = dA[index];
else
dZ[index] = 0;
}
}