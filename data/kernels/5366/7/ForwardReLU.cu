#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardReLU(float* Z, int nRowsZ, int nColsZ, float* A)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < nRowsZ * nColsZ)
{
if (Z[index] >= 0)
A[index] = Z[index];
else
A[index] = 0;
}
}