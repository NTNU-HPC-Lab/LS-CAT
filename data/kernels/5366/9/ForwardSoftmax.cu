#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardSoftmax(float *Z, int nColsZ, float *sumExp, float *A)
{
int row = threadIdx.x;
int col = blockIdx.x;

atomicAdd(&sumExp[col], exp(Z[row * nColsZ + col]));

__syncthreads();

A[row * nColsZ + col] = exp(Z[row * nColsZ + col]) / sumExp[col];
}