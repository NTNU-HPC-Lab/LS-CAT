#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardLinear(float *A, float *W, float *b, int nRowsW, int nColsW, int nColsA, float *Z)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

float ZValue = 0;

if (row < nRowsW && col < nColsA)
{
for (int i = 0; i < nColsW; i++)
{
ZValue += W[row * nColsW + i] * A[i * nColsA + col];
}
Z[row * nColsA + col] = ZValue + b[row];
}
}