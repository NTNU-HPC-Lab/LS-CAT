#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UpdateParamsLinear(float *dZ, float *A, int nRowsdZ, int nColsdZ, int nRowsA, float lr, float *W, float *b)
{
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

float dWValue = 0, dbValue = 0;

if (row < nRowsdZ && col < nRowsA)
{
for (int i = 0; i < nColsdZ; i++)
{
dWValue += dZ[row * nColsdZ + i] * A[col * nColsdZ + i];
}
W[row * nRowsA + col] = W[row * nRowsA + col] - lr * dWValue / nColsdZ;

if (col == 0)
{
for (int i = 0; i < nColsdZ; i++)
{
dbValue += dZ[row * nColsdZ + i];
}
b[row] = b[row] - lr * dbValue / nColsdZ;
}
}
}