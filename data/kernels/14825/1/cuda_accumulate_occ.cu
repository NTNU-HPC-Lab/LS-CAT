#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_accumulate_occ(float * device_mapOcc, int numObjs, int numClusters, int clusterStart, int sub_numClusters,  float *device_reduceOcc) {
int objIndex = blockDim.x * blockIdx.x + threadIdx.x;
extern __shared__ float shared_objects[];

if(objIndex < numObjs) {
for(int i = 0;i < sub_numClusters; i++)
//        for(int i = clusterStart + sub_numClusters - 1;i >= clusterStart; i--)
shared_objects[threadIdx.x * sub_numClusters + i] = device_mapOcc[objIndex * numClusters + i + clusterStart];
}
else {
for(int i = 0;i < sub_numClusters; i++)
//for(int i = clusterStart + sub_numClusters - 1;i >= clusterStart; i--)
shared_objects[threadIdx.x * sub_numClusters + i] = 0;
}

__syncthreads();

for(int i = (blockDim.x >> 1); i >= 1; i>>=1) {
if(threadIdx.x < i) {
for(int j = 0;j < sub_numClusters; j++) {
//for(int j = clusterStart + sub_numClusters - 1;j >= clusterStart; j--)
shared_objects[threadIdx.x * sub_numClusters + j] += shared_objects[(threadIdx.x + i) * sub_numClusters + j];
}
}
__syncthreads();
}

if(threadIdx.x == 0) {
for(int i = 0;i < sub_numClusters;i++) {
//        for(int i = clusterStart + sub_numClusters - 1;i >= clusterStart; i--)
device_reduceOcc[blockIdx.x * numClusters + i + clusterStart]  = shared_objects[i];
}
}
}