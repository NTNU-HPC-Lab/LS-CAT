#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_packed_cl_cs(int *d_packed_cl, int *d_packed_cs, int *d_cl, int *d_cs, int *d_gcs, int chunk_num)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= chunk_num) {
return;
}

if (d_gcs[i + 1] - d_gcs[i] > 0) {
d_packed_cl[d_gcs[i]] = d_cl[i];
d_packed_cs[d_gcs[i]] = d_cs[i];
}
}