#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_cs(int *d_cl, int *d_cs, int c_size, int chunk)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= c_size) {
return;
}

if (i == 0) {
d_cs[i] = 0;
}
else {
d_cs[i] = d_cl[i - 1] * chunk;
}

}