#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= M) {
return;
}
int nz_per_row = d_row_nz[i];

atomicMax(d_max, nz_per_row);

int j = 0;
for (j = 0; j < BIN_NUM - 2; j++) {
if (nz_per_row <= (min << j)) {
if (nz_per_row <= (mmin)) {
atomicAdd(d_bin_size + j, 1);
}
else {
atomicAdd(d_bin_size + j + 1, 1);
}
return;
}
}
atomicAdd(d_bin_size + BIN_NUM - 1, 1);
}