#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zero_fill_int(int *d_array, int size) {

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i >= size) {
return;
}

d_array[i] = 0;

}