#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_c_size(int *d_c_size, int *d_full_cl, int size)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i >= size) {
return;
}

if (d_full_cl[i] != 0) {
atomicAdd(d_c_size, 1);
}
}