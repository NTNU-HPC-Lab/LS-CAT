#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCalculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
int id = blockDim.x * blockIdx.x + threadIdx.x;

int startPosition = id * chunkSize;
for (int i = startPosition; i < (startPosition + chunkSize); i++) {
if (i < totalPixels) {
int pixelValue = (int)rawPixels[i];
atomicAdd(&histogram[pixelValue], 1);
}
}
}