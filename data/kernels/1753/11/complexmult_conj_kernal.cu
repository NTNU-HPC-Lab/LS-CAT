#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void complexmult_conj_kernal(float *afft, const float *bfft, int totaltc)
{

const uint ridx = 2*(threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS);

//maybe use float2 to improve coalessing....
if (ridx < totaltc){
const uint iidx = ridx + 1;
float afftr = afft[ridx];
float affti = afft[iidx];
float bfftr = bfft[ridx];
float bffti = bfft[iidx];

afft[ridx] = afftr*bfftr + affti*bffti;  //real portion
afft[iidx] = affti*bfftr - afftr*bffti; //imaginary portion
}

}