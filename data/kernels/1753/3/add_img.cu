#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_img(float *image_padded, float *ave1, float *ave2, int nx, int ny, int nima) {

// Block index
int bx = blockIdx.x;

// Thread index
int tx = threadIdx.x;

float sum1 = 0.0;
float sum2 = 0.0;
int index = tx+bx*nx;
int index2 = tx+(nx>>1)+(bx+(ny>>1))*(nx*2+2);

for (int i=0; i<nima; i+=2) sum1 += image_padded[index2+i*(nx*2+2)*ny*2];
for (int i=1; i<nima; i+=2) sum2 += image_padded[index2+i*(nx*2+2)*ny*2];
ave1[index] = sum1;
ave2[index] = sum2;

return;
}