#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ccc_cmp_kernal(const float* data1, const float* data2, float* device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{
float avg1 = 0.0f;
float avg2 = 0.0f;
float var1 = 0.0f;
float var2 = 0.0f;
float ccc = 0.0f;

const uint x=threadIdx.x;
const uint y=blockIdx.x;

int idx = x + y*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
avg1 += data1[index];
avg2 += data2[index];
var1 += data1[index]*data1[index];
var2 += data2[index]*data2[index];
ccc += data1[index]*data2[index];
}

device_soln[idx] = avg1/size;
device_soln[idx + num_calcs] = avg2/size;
device_soln[idx + 2*num_calcs] = var1/size;
device_soln[idx + 3*num_calcs] = var2/size;
device_soln[idx + 4*num_calcs] = ccc/size;


}