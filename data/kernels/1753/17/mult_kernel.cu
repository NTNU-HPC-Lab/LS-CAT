#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_kernel(float* data, const float scale, const int realtc)
{

const uint index = threadIdx.x + (blockIdx.x + gridDim.x*blockIdx.y)*MAX_THREADS;

if (index < realtc){
data[index] *= scale;
}
}