#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swap_middle_row(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
const uint x=threadIdx.x;
const uint y=blockIdx.x;

const uint c = x+y*num_threads+offset;
int r = ny/2;
int idx1 = r*nx + c;
int idx2 = r*nx + c + nx/2+ xodd;
float tmp = data[idx1];
data[idx1] = data[idx2];
data[idx2] = tmp;
}