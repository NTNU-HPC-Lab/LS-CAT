#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_cmp_kernal(const float* data1, const float* data2, float* device_soln, const int size, const int num_threads, const int offset)
{
float dot = 0.0f;

int idx = threadIdx.x + blockIdx.x*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
dot += data1[index]*data2[index];
}

device_soln[idx] = dot/size;

}