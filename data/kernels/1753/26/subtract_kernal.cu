#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtract_kernal(float* data, float f, const int totaltc)
{

int idx = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS;

if(idx < totaltc){
data[idx] = data[idx] - f;
}
}