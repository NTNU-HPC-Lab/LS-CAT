#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swap_middle_column(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
const uint x=threadIdx.x;
const uint y=blockIdx.x;

const uint r = x+y*num_threads+offset;
int c = nx/2;
int idx1 = r*nx + c;
int idx2 = (r+ny/2+yodd)*nx + c;
float tmp = data[idx1];
data[idx1] = data[idx2];
data[idx2] = tmp;
}