#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void norm_kernal(float * data, float mean, float var, int totaltc)
{

const uint index = threadIdx.x + (blockIdx.x + gridDim.x*blockIdx.y)*MAX_THREADS;

if(index < totaltc){
data[index] = (data[index] - mean)/var;
}

}