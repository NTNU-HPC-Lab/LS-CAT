#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void middle_to_top(float* data, const int nx, const int ny)
{
float tmp;
for ( int c = 0; c < nx; ++c ) {
// Get the value in the top row
float last_val = data[ny/2*nx + c];
for ( int r = ny-1; r >= ny/2; --r ){
int idx = r*nx+c;
tmp = data[idx];
data[idx] = last_val;
last_val = tmp;
}
}
}