#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumup_kernal(const float * data, float * device_stats, const int size, const int dim2size, const int num_threads, const int offset)
{
float sum = 0.0f;

const uint x=threadIdx.x;
const uint y=blockIdx.x;

int idx = x + y*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*dim2size + idx % dim2size;
sum += data[index];
}

device_stats[idx] = sum/size;
}