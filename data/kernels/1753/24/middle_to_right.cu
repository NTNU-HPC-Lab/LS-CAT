#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void middle_to_right(float* data, const int nx, const int ny)
{
float tmp;
for ( int r  = 0; r < ny; ++r ) {
float last_val = data[r*nx+nx/2];
for ( int c = nx-1; c >=  nx/2; --c ){
int idx = r*nx+c;
tmp = data[idx];
data[idx] = last_val;
last_val = tmp;
}
}
}