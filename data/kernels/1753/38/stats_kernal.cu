#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stats_kernal(const float *data, float * device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{

float sum = 0.0f;
float sum_sq = 0.0f;

int idx = threadIdx.x + blockIdx.x*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
float datum = data[index]; //so we dno't need multiple accesses to global mem, I would think that the compiler would optimize this, but the manual said to program like this....
sum += datum;
sum_sq += datum*datum;
}

device_soln[idx] = sum;
device_soln[idx + num_calcs] = sum_sq;

}