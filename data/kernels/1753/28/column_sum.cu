#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void column_sum(const float* data, float* sum, int nx, int ny, int num_threads, int offset ) {

float s = 0.0;
const uint idx = threadIdx.x + blockIdx.x*num_threads+offset;
for(int i =0; i < ny; i++) {
s += data[idx + i*nx];
}
sum[idx] = s;
}