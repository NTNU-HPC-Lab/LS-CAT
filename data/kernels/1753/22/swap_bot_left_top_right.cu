#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swap_bot_left_top_right(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
const uint x=threadIdx.x;
const uint y=blockIdx.x;

const uint gpu_idx = x+y*num_threads+offset;
const uint c = gpu_idx % (nx/2);
const uint r = gpu_idx / (nx/2);

const uint idx1 = r*nx + c;
const uint idx2 = (r+ny/2+yodd)*nx + c + nx/2+xodd;
float tmp = data[idx1];
data[idx1] = data[idx2];
data[idx2] = tmp;
}