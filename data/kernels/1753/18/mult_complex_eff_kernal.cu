#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_complex_eff_kernal(float* data, const float* src_data, const int nx, const int nxy, const int size)
{
int idx = threadIdx.z*nxy + threadIdx.y*nx + threadIdx.x;

data[idx] *= src_data[idx];
data[size-idx-1] *= src_data[size-idx-1];
}