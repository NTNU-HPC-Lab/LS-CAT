#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tovalue_kernal(float* data, const float value, const int totaltc)
{

const uint idx = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS;

if(idx < totaltc){
data[idx] = value;
}

}