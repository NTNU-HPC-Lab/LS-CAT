#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_cmp_kernal_reduce(float *g_idata1, float *g_idata2, float *g_odata)
{
extern __shared__ float sdata[];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata1[i]*g_idata2[i] + g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}