#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_180( float* data,int nx, int nxy, int offset, unsigned int size) {

const uint x=threadIdx.x;
const uint y=blockIdx.x;

__shared__ float shared_lower_data[MAX_THREADS];
__shared__ float shared_upper_data[MAX_THREADS];

shared_lower_data[x] = data[x+y*MAX_THREADS+offset];
shared_upper_data[x] = data[nxy + x+(-y-1)*MAX_THREADS-offset];
__syncthreads();


if (size == 0) {
float tmp = shared_lower_data[x];
shared_lower_data[x] = shared_upper_data[MAX_THREADS-x-1];
shared_upper_data[MAX_THREADS-x-1] = tmp;
} else {
if ( x < size ) {
float tmp = shared_lower_data[x];
shared_lower_data[x] = shared_upper_data[MAX_THREADS-x-1];
shared_upper_data[MAX_THREADS-x-1]= tmp;

}
}

__syncthreads();
if (size == 0) {
data[x+y*MAX_THREADS+offset] = shared_lower_data[x];
data[nxy+x+(-y-1)*MAX_THREADS-offset] = shared_upper_data[x];
} else {
if ( x < size ) {
data[nxy-x-1+(-y)*MAX_THREADS-offset] = shared_upper_data[MAX_THREADS-x-1];
data[x+y*MAX_THREADS+offset] = shared_lower_data[x];
}
}

}