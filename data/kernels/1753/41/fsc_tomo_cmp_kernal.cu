#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fsc_tomo_cmp_kernal(const float* data1, const float* data2, float* device_soln, const float data1threshold, const float data2threshold, const int nx, const int ny, const int nz, const int offset)
{

const uint x=threadIdx.x;
const uint y=blockIdx.x;

int idx = x + y*MAX_THREADS + offset;

float sum_data1_amps = 0.0;
float sum_data2_amps = 0.0;
float top = 0.0;
for(int i = 0; i < ny; i++){
//int index = i*nx + idx % nx + ((idx/nx)*ny*nz); //for coalesing
int rindex = i*nx + 2*(idx % nx/2) + (2*idx/nx)*ny*nz;
int iindex = i*nx + 2*(idx % nx/2)+ 1 + (2*idx/nx)*ny*nz;
float data1_r = data1[rindex];
float data1_i = data1[iindex];
float data2_r = data2[rindex];
float data2_i = data2[iindex];
if((data1_r* data1_r +  data1_i*data1_i) > data1threshold && (data2_r* data2_r +  data2_i*data2_i) > data2threshold){
sum_data1_amps += (data1_r* data1_r +  data1_i*data1_i);
sum_data2_amps += (data2_r* data2_r +  data2_i*data2_i);
top += (data1_r*data2_r + data1_i*data2_i);
}
}
device_soln[idx*3] = top;
device_soln[idx*3 +1] = sum_data1_amps;
device_soln[idx*3 +2] = sum_data2_amps;

}