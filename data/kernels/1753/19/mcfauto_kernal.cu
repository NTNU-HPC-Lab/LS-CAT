#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mcfauto_kernal(const float* data1, float* data2, const int totaltc)
{
int idx = 2*(threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS);

if(idx < totaltc){
data2[idx] = sqrt(data1[idx] * data2[idx] + data1[idx + 1] * data2[idx + 1]);
data2[idx + 1] = 0;
}
}