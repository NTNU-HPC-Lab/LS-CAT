#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_edgemean_kernal(const float* data, float* edgemean, const int nx, const int ny, const int nz)
{
int di = 0;
float edge_sum = 0;
float edge_mean = 0;
size_t nxy = nx * ny;
if (nz == 1) {
for (int i = 0, j = (ny - 1) * nx; i < nx; ++i, ++j) {
edge_sum += data[i] + data[j];
}
for (size_t i = 0, j = nx - 1; i < nxy; i += nx, j += nx) {
edge_sum += data[i] + data[j];
}
edge_mean = (float)edge_sum / (nx * 2 + ny * 2);
}
else {
if (nx == ny && nx == nz * 2 - 1) {
for (size_t j = (nxy * (nz - 1)); j < nxy * nz; ++j, ++di) {
edge_sum += data[j];
}
}
else {
for (size_t i = 0, j = (nxy * (nz - 1)); i < nxy; ++i, ++j, ++di) {
edge_sum += data[i] + data[j];
}
}

int nxy2 = nx * (ny - 1);
for (int k = 1; k < nz - 1; ++k) {
size_t k2 = k * nxy;
size_t k3 = k2 + nxy2;
for (int i = 0; i < nx; ++i, ++di) {
edge_sum += data[i + k2] + data[i + k3];
}
}
for (int k = 1; k < nz - 1; ++k) {
size_t k2 = k * nxy;
size_t k3 = nx - 1 + k2;
for (int i = 1; i < ny - 1; ++i, ++di) {
edge_sum += data[i * nx + k2] + data[i * nx + k3];
}
}

edge_mean = (float)edge_sum / (di * 2);
}
*edgemean = edge_mean;
}