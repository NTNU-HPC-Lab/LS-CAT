#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernal_reduce(float *g_idata, float *g_odata)
{
extern __shared__ float sdata[];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}