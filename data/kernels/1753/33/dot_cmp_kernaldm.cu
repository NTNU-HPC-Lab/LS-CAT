#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_cmp_kernaldm(const float* data1, const float* data2, const float* dm, float* device_soln, const int size, const int num_threads, const int offset)
{
float dot = 0.0f;
float nnn = 0.0f;

int idx = threadIdx.x + blockIdx.x*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
if(dm[index] > 0.5){
dot += data1[index]*data2[index];
nnn += 1.0f;
}
}

device_soln[idx] = dot/nnn;

}