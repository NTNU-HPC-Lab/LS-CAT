#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernal(const float * data, float * device_stats, const int size, const int num_calcs, const int num_threads, const int offset)
{
float mean = 0.0f;

const uint x=threadIdx.x;
const uint y=blockIdx.x;
int idx = x + y*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
mean += data[index];
}

device_stats[idx] = mean/size;
}