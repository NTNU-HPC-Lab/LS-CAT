#include "hip/hip_runtime.h"
#include "includes.h"
// Includes, system

// prototype function
//int rnd_asg(unsigned short int*, unsigned int*, int, int);
//void criterion_part(float*, unsigned short int*, unsigned int*, float*, float*, float*, float*, int, int, int);

// ERROR system
#define EXIT_OK (0)
#define ERROR_HOST_MEM (1)
#define ERROR_DEVICE_MEM (2)
#define ERROR_DEVICE (3)
#define ERROR_INIT (4)
#define ERROR_EMPTY (5)
#define ERROR_SETDEVICE (6)
#define EXIT_DONE (255)

// kernel to calculate the exp

__global__ void kmeans_exp_kernel(float* DIST, float pw)
{
register int idx = blockIdx.x * blockDim.x + threadIdx.x;
register float arg = DIST[idx] * pw;
if (arg < -70) arg = -70;
DIST[idx] = exp(arg);
}