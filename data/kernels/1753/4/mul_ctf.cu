#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_ctf(float *image, int nx, int ny, float defocus, float cs, float voltage, float apix, float bfactor, float ampcont) {

// Block index
int bx = blockIdx.x;

// Thread index
int tx = threadIdx.x;

float x, y;

x = float(bx);
if (tx >= ny>>1) y = float(tx-ny);
else y = float(tx);
int index = bx*2+tx*(nx+2);

float ak = sqrt(x*x+y*y)/nx/apix;
float cst = cs*1.0e7f;
float wgh = ampcont/100.0f;
float phase = atan(wgh/sqrt(1.0f-wgh*wgh));
float lambda = 12.398f/sqrt(voltage*(1022.f+voltage));
float ak2 = ak*ak;
float g1 = defocus*1.0e4f*lambda*ak2;
float g2 = cst*lambda*lambda*lambda*ak2*ak2/2.0f;
float ctfv = sin(PI*(g1-g2)+phase);
if (bfactor != 0.0f)  ctfv *= exp(-bfactor*ak2/4.0f);

image[index] *= ctfv;
image[index+1] *= ctfv;
}