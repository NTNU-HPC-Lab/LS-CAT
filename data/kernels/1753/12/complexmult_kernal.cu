#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void complexmult_kernal(float *afft, const float *bfft, int totaltc)
{

const uint ridx = 2*(threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS);

if(ridx < totaltc){
const uint iidx = ridx + 1;
//maybe use float2 to improve coalessing....

float afftr = afft[ridx];
float affti = afft[iidx];
float bfftr = bfft[ridx];
float bffti = bfft[iidx];

afft[ridx] = afftr*bfftr - affti*bffti;  //real portion
afft[iidx] = affti*bfftr + afftr*bffti; //imaginary portion
}

}