#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dsa_sumup_kernal(float* results, const int inx, const int iny)
{
int tidx = threadIdx.x;
int bd = blockDim.x;
int size = iny*(inx/2 + 1);

float dot = 0.0f; float vweight = 0.0f; float power = 0.0f; float power2 = 0.0f;
for (int i = 0; i < (inx/2 + 1); i++) {
int idx = i*bd + tidx;
dot += results[idx];
vweight += results[size + idx];
power += results[2*size + idx];
power2 += results[3*size + idx];
}

results[tidx] = dot;
results[size + tidx] = vweight;
results[2*size + tidx] = power;
results[3*size + tidx] = power2;

}