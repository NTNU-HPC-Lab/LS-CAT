#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ccc_cmp_kernaldm(const float* data1, const float* data2, const float* dm, float* device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{
float avg1 = 0.0f;
float avg2 = 0.0f;
float var1 = 0.0f;
float var2 = 0.0f;
float ccc = 0.0f;
float nnn = 0.0f;

const uint x=threadIdx.x;
const uint y=blockIdx.x;

int idx = x + y*num_threads + offset;

for(int i = 0; i < size; i++){
int index = i*size + idx % size + ((idx/size)*size*size); //for coalesing
if(dm[index] > 0.5){
avg1 += data1[index];
avg2 += data2[index];
var1 += data1[index]*data1[index];
var2 += data2[index]*data2[index];
ccc += data1[index]*data2[index];
nnn += 1.0f;
}
}

device_soln[idx] = avg1/nnn;
device_soln[idx + num_calcs] = avg2/nnn;
device_soln[idx + 2*num_calcs] = var1/nnn;
device_soln[idx + 3*num_calcs] = var2/nnn;
device_soln[idx + 4*num_calcs] = ccc/nnn;


}