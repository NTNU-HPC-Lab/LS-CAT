#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_kernel(float* DIST, float pw)
{
register int idx = blockIdx.x * blockDim.x + threadIdx.x;
register float arg = DIST[idx] * pw;
if (arg < -70) arg = -70;
DIST[idx] = exp(arg);
}