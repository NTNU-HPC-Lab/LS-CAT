#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREADS 	743511 	// length of calculation
#define BLOCK_SIZE 	256	// number of threads per block used in gpu calc
#define EPS		0.00005 // Epsilon for tolerance of diffs between cpu and gpu calculations
#define INCLUDE_MEMTIME false	// Decides whether to include memory transfers to and from gpu in gpu timing
#define PRINTLINES	0	// Number of lines to print in output during validation



int timeval_subtract(  struct timeval* result,
struct timeval* t2,
struct timeval* t1) {
unsigned int resolution = 1000000;
long int diff = (t2->tv_usec + resolution * t2->tv_sec) -
(t1->tv_usec + resolution * t1->tv_sec);
result->tv_sec = diff / resolution;
result->tv_usec = diff % resolution;
return (diff<0);
}

__global__ void calcKernel(float* d_in, float *d_out) {
const unsigned int lid = threadIdx.x;			// local id inside a block
const unsigned int gid = blockIdx.x*blockDim.x + lid; 	// global id
d_out[gid] = pow((d_in[gid] / ( d_in[gid] - 2.3 )),3);	// do computation
}