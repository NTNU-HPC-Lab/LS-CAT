#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/




#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
return i * MAX_CITIES + j;
}
__global__ void updateTrailsAtomic(float *phero, int *paths, float *tourLengths)
{
int antId = blockIdx.x;
int from, to;

for (int i = 0; i < MAX_CITIES; i++) {
from = paths[toIndex(antId, i)];
if (i < MAX_CITIES - 1) {
to = paths[toIndex(antId, i+1)];
} else {
to = paths[toIndex(antId, 0)];
}

if (from < to) {
int tmp = from;
from = to;
to = tmp;
}
atomicAdd(&phero[toIndex(from, to)], QVAL / tourLengths[antId]);
}
}