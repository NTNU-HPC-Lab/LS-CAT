#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/




#define MAX_THREADS 128

__device__ static inline int calculateTo(int i) {
//find least triangle number less than i
int row = (int)(-1 + (sqrt((float)(1 + 8 * i)))) >> 1;
int tnum = (row * (row + 1)) >> 1;
int remain = i - tnum;
return row - remain;
}
__device__ static inline int calculateFrom(int i) {
//find least triangle number less than i
int row = (int)(-1 + (sqrt((float)(1 + 8 * i)))) >> 1;
int tnum = (row * (row + 1)) >> 1;
int remain = i - tnum;
return MAX_CITIES - 1 - remain;
}
__device__ static inline int toIndex(int i, int j) {
return i * MAX_CITIES + j;
}
__global__ void updateTrails(float *phero, int *paths, float *tourLengths)
{
//__shared__ float localPaths[MAX_CITIES];

int numPhero = (NUM_EDGES + (blockDim.x * (MAX_ANTS * 2) - 1)) /
(blockDim.x * (MAX_ANTS * 2));
int blockStartPhero = numPhero * blockDim.x * blockIdx.x;
int from, to;

int cur_phero;
for (int i = 0; i < MAX_ANTS; i++) {
// For each ant, cache paths in shared memory
/*int tile;
if (startCityIndex + citiesPerThread >= MAX_CITIES) {
tile = MAX_CITIES - startCityIndex;
} else {
tile = citiesPerThread;
}
memcpy(&localPaths[startCityIndex], &paths[i * MAX_CITIES + startCityIndex], tile * sizeof(float));
*/
// TODO: figure out tiling
/*if (threadIdx.x == 0) {
memcpy(&localPaths, &paths[i * MAX_CITIES], MAX_CITIES * sizeof(float));
}

__syncthreads();
*/

for (int j = 0; j < numPhero; j++) {
cur_phero = blockStartPhero + j + numPhero * threadIdx.x;

if (cur_phero >= NUM_EDGES) {
break;
}

from = calculateFrom(cur_phero); //triangle number thing
to = calculateTo(cur_phero);

bool touched = false;
int checkTo;
int checkFrom;
for (int k = 0; k < MAX_CITIES; k++) {
checkFrom = paths[toIndex(i, k)];
if (k < MAX_CITIES - 1) {
checkTo = paths[toIndex(i, k + 1)];
} else {
checkTo = paths[toIndex(i, 0)];
}

if ((checkFrom == from && checkTo == to) ||
(checkFrom == to && checkTo == from))
{
touched = true;
break;
}
}

if (touched) {
int idx = toIndex(from, to);
phero[idx] += (QVAL / tourLengths[i]);
phero[toIndex(to, from)] = phero[idx];
}
}
//__syncthreads();
}
}