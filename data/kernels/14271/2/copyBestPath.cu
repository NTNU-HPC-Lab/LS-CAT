#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/




#define MAX_THREADS 128

__global__ void copyBestPath(int i, int *bestPathResult, int *pathResults) {
memcpy(bestPathResult, &pathResults[i * MAX_ANTS], MAX_CITIES * sizeof(int));
}