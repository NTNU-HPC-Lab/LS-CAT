#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/




#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
return i * MAX_CITIES + j;
}
__global__ void seqPheroUpdate(float *phero, float *pheroReal, int *paths, float *tourLengths) {
memcpy(phero, pheroReal, sizeof(float) * MAX_CITIES * MAX_CITIES);

int from, to;
// evaporate
for (from = 0; from < MAX_CITIES; from++) {
for (to = 0; to < from; to++) {
phero[toIndex(from, to)] *= 1.0 - RHO;

if (phero[toIndex(from, to)] < 0.0) {
phero[toIndex(from, to)] = INIT_PHER;
}
phero[toIndex(to, from)] = phero[toIndex(from, to)];
}
}

//Add new pheromone to the trails
for (int ant = 0; ant < MAX_ANTS; ant++) {
for (int i = 0; i < MAX_CITIES; i++) {
from = paths[toIndex(ant, i)];
if (i < MAX_CITIES - 1) {
to = paths[toIndex(ant, i+1)];
} else {
to = paths[toIndex(ant, 0)];
}

phero[toIndex(from, to)] += (QVAL / tourLengths[ant]);
phero[toIndex(to, from)] = phero[toIndex(from, to)];
}
}

}