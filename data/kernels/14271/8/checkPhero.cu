#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/




#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
return i * MAX_CITIES + j;
}
__global__ void checkPhero(float *pheroSeq, float *phero) {
for (int i = 0; i < MAX_CITIES; i++) {
for (int j = 0; j < MAX_CITIES; j++) {
if (i == j) continue;
int idx = toIndex(i, j);
if (fabsf(pheroSeq[idx] - phero[idx]) > 0.001) {
printf("PHERO IS BROKEN at (%d, %d); expected: %1.15f, actual: %1.15f\n", i, j, pheroSeq[idx], phero[idx]);
}
}
}
}