#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void create_fpr_kernel(float* tpr, const int* unique_index, float* fpr, int num_selected, int num_total) {
float pos_cnt = tpr[num_selected - 1];
float neg_cnt = num_total - pos_cnt;
int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
for (int gid = gid_base; gid < num_selected; gid += blockDim.x * gridDim.x) {
float tp = tpr[gid];
fpr[gid] = (1.0f + unique_index[gid] - tp) / neg_cnt;
tpr[gid] = tp / pos_cnt;
}
}