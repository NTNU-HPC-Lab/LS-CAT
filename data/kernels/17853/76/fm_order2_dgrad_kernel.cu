#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fm_order2_dgrad_kernel(const float* in, const float* top_grad, float* dgrad, int batch_size, int slot_num, int emb_vec_size) {
int tid = threadIdx.x;
int bid = blockIdx.x;

if (tid < emb_vec_size && bid < batch_size) {
float emb_sum = 0.0f;
int offset = bid * slot_num * emb_vec_size + tid;

for (int i = 0; i < slot_num; i++) {
int index = offset + i * emb_vec_size;
emb_sum += in[index];
}
float tgrad = top_grad[bid * emb_vec_size + tid];
for (int i = 0; i < slot_num; i++) {
int index = offset + i * emb_vec_size;
dgrad[index] = tgrad * (emb_sum - in[index]);
}
}
}