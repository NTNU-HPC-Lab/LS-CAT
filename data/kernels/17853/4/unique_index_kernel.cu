#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unique_index_kernel(const char* flag, const int* flag_inc_sum, int* unique_index, int num_elems) {
int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
for (int gid = gid_base; gid < num_elems; gid += blockDim.x * gridDim.x) {
if (flag[gid] == 1) {
int id = flag_inc_sum[gid] - 1;
unique_index[id] = gid;
}
}
}