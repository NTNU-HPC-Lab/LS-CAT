#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size, const size_t *deltaw_hash_value_index, const float *deltaw, float *hash_table_value) {
int tid = threadIdx.x;
int bid = blockIdx.x;

if ((bid < hash_value_index_count_num) && (tid < embedding_vec_size)) {
size_t value_index = deltaw_hash_value_index[bid];
size_t feature_index = value_index * embedding_vec_size + tid;
hash_table_value[feature_index] += deltaw[bid * embedding_vec_size + tid];
}
}