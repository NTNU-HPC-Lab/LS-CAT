#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void value_count_kernel_1(int nnz, const size_t *hash_value_index_sort, uint32_t *new_hash_value_flag) {
for (int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < nnz; gid += blockDim.x * gridDim.x) {
size_t cur_value = hash_value_index_sort[gid];
if (gid > 0) {
size_t former_value = hash_value_index_sort[gid - 1];
// decide if this is the start of a group(the elements in this group have the same
// hash_value_index_sort)
if (cur_value != former_value) {
new_hash_value_flag[gid] = 1;
} else {
new_hash_value_flag[gid] = 0;
}
} else {  // gid == 0
new_hash_value_flag[gid] = 1;
}
}
}