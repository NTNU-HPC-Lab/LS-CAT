#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fm_order2_kernel(const float* in, float* out, int batch_size, int slot_num, int emb_vec_size) {
int tid = threadIdx.x;
int bid = blockIdx.x;

if (tid < emb_vec_size && bid < batch_size) {
float emb_sum = 0.0f;
float emb_sum_square = 0.0f;
float emb_square_sum = 0.0f;
int offset = bid * slot_num * emb_vec_size + tid;

for (int i = 0; i < slot_num; i++) {
int index = offset + i * emb_vec_size;
float temp = in[index];
emb_sum += temp;
emb_square_sum += temp * temp;
}
emb_sum_square = emb_sum * emb_sum;

out[bid * emb_vec_size + tid] = 0.5f * (emb_sum_square - emb_square_sum);
}
}