#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void trapz_kernel(float* y, float* x, float* auc, int num_selected) {
__shared__ float s_auc;
s_auc = 0.0f;
__syncthreads();
int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
for (int gid = gid_base; gid < num_selected - 1; gid += blockDim.x * gridDim.x) {
float a = x[gid];
float b = x[gid + 1];
float fa = y[gid];
float fb = y[gid + 1];
float area = (b - a) * (fa + fb) / 2.0f;
if (gid == 0) {
area += (a * fa / 2.0f);
}
atomicAdd(&s_auc, area);
}
__syncthreads();
if (threadIdx.x == 0) {
atomicAdd(auc, s_auc);
}
}