#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaxpy(double *y, double *a, double *x, int m, int n){
int bid = blockIdx.x;
int tid = threadIdx.x;
extern __shared__ double dots_s[];
if(bid<m)
if(tid<n){

dots_s[bid*n+tid] = a[bid*n+tid] * *(x+tid);
__syncthreads();
if(tid == 0){
for(int i=1;i<n;i++){
dots_s[bid*n] +=dots_s[bid*n+i];
//			printf("y=%d, dots_s=%d, bid=%d, tid=%d, i=%d, n=%d\n",dots_s[bid*n], dots_s[bid*n+i],bid,tid,i,n);
}
*(y+bid)=dots_s[bid*n];
//		printf("y[%d]=%d, bid=%d, tid=%d\n",bid,y[bid],bid,tid);
}
}
}