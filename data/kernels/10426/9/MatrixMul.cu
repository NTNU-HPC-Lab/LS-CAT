#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
// calculate thread id
unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

for (int k = 0 ; k<WIDTH ; k++ )
{
Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col];
}
}