#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(int * _matrixA, int * _matrixB, int * _result, int _width)
{
int k = 0, elementA = 0, elementB = 0;
//2D thread ID
int tx = threadIdx.x;
int ty = threadIdx.y;

//valeu store the _result element that is computed by thread
int value = 0;
for (k = 0; k < _width; k++)
{
elementA = *(_matrixA + (ty*_width + k));  //Go accross the line
elementB = *(_matrixB + (k*_width + tx));  //Go accross the column
value += (elementA * elementB);   //Take each element
}
*(_result + (_width*ty + tx)) = value;

return;
}