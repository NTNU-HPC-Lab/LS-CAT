#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_shmem ( float* y, float* x, float a, clock_t * timer_vals)
{
volatile __shared__ float sdata_x0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y0 [COMPUTE_THREADS_PER_CTA];
int tid = threadIdx.x ;
for (int i=0; i < NUM_ITERS; ++i) {
unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
__syncthreads();
sdata_x0[tid] = x[idx];
sdata_y0[tid] = y[idx];
__syncthreads();
y[idx] = a * sdata_x0[tid] + sdata_y0[tid];
}
}