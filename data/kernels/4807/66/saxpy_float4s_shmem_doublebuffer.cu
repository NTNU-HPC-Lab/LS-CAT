#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s_shmem_doublebuffer ( float* y, float* x, float a, clock_t * timer_vals)
{
volatile __shared__ float sdata_x0_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x1_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x2_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x3_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y0_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y1_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y2_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y3_0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x0_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x1_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x2_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x3_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y0_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y1_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y2_1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y3_1 [COMPUTE_THREADS_PER_CTA];
int tid = threadIdx.x ;

unsigned int idx0, idx1;
idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;

float4 * x_as_float4 = (float4 *)x;
float4 * y_as_float4 = (float4 *)y;
float4 result_y;

for (int i=0; i < NUM_ITERS/4; i+=2) {
float4 tmp1_x, tmp1_y;

__syncthreads();
tmp1_x = x_as_float4[idx0];
tmp1_y = y_as_float4[idx0];
if (i!=0) {
result_y.x = a * sdata_x0_1[tid] + sdata_y0_1[tid];
result_y.y = a * sdata_x1_1[tid] + sdata_y1_1[tid];
result_y.z = a * sdata_x2_1[tid] + sdata_y2_1[tid];
result_y.w = a * sdata_x3_1[tid] + sdata_y3_1[tid];
y_as_float4[idx1] = result_y;
idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT ;
}
sdata_x0_0[tid] = tmp1_x.x;
sdata_x1_0[tid] = tmp1_x.y;
sdata_x2_0[tid] = tmp1_x.z;
sdata_x3_0[tid] = tmp1_x.w;
sdata_y0_0[tid] = tmp1_y.x;
sdata_y1_0[tid] = tmp1_y.y;
sdata_y2_0[tid] = tmp1_y.z;
sdata_y3_0[tid] = tmp1_y.w;
__syncthreads();
tmp1_x = x_as_float4[idx1];
tmp1_y = y_as_float4[idx1];
result_y.x = a * sdata_x0_0[tid] + sdata_y0_0[tid];
result_y.y = a * sdata_x1_0[tid] + sdata_y1_0[tid];
result_y.z = a * sdata_x2_0[tid] + sdata_y2_0[tid];
result_y.w = a * sdata_x3_0[tid] + sdata_y3_0[tid];
y_as_float4[idx0] = result_y;
idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT ;
sdata_x0_1[tid] = tmp1_x.x;
sdata_x1_1[tid] = tmp1_x.y;
sdata_x2_1[tid] = tmp1_x.z;
sdata_x3_1[tid] = tmp1_x.w;
sdata_y0_1[tid] = tmp1_y.x;
sdata_y1_1[tid] = tmp1_y.y;
sdata_y2_1[tid] = tmp1_y.z;
sdata_y3_1[tid] = tmp1_y.w;
}
__syncthreads();
result_y.x = a * sdata_x0_1[tid] + sdata_y0_1[tid];
result_y.y = a * sdata_x1_1[tid] + sdata_y1_1[tid];
result_y.z = a * sdata_x2_1[tid] + sdata_y2_1[tid];
result_y.w = a * sdata_x3_1[tid] + sdata_y3_1[tid];
y_as_float4[idx1] = result_y;

}