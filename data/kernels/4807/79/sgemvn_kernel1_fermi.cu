#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemvn_kernel1_fermi(int n, int m, int n1, float alpha, float* A, int lda, float *x, float *y)
{
int ind = blockIdx.x*num_threads + threadIdx.x;

A += ind;

float res = 0.f;

for(int i=0; i<n1; i += sgemv_bs ){

#pragma unroll
for(int j=0; j < sgemv_bs ; j++){
res += A[0] * x[j];
A   += lda;
}
x += sgemv_bs;
}

#if 0
if (m>n1){

for(int j=0; j<(m-n1); j++){
res += A[0] * x[j];
A   += lda;
}
}
#endif

if (ind<n)
y[ind] = alpha * res;

}