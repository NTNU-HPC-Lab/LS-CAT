#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_baseline ( float* y, float* x, float a, clock_t * timer_vals)
{
for (int i=0; i < NUM_ITERS; i++) {
unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
y[idx] = a * x[idx] + y[idx];
}
}