#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s_shmem ( float* y, float* x, float a, clock_t * timer_vals)
{
volatile __shared__ float sdata_x0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x2 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_x3 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y0 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y1 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y2 [COMPUTE_THREADS_PER_CTA];
volatile __shared__ float sdata_y3 [COMPUTE_THREADS_PER_CTA];
int tid = threadIdx.x ;

for (int i=0; i < NUM_ITERS/4; i++) {
unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;

__syncthreads();
float4 * x_as_float4 = (float4 *)x;
float4 * y_as_float4 = (float4 *)y;
float4 tmp1_x, tmp1_y;
tmp1_x = x_as_float4[idx];
tmp1_y = y_as_float4[idx];
sdata_x0[tid] = tmp1_x.x;
sdata_x1[tid] = tmp1_x.y;
sdata_x2[tid] = tmp1_x.z;
sdata_x3[tid] = tmp1_x.w;
sdata_y0[tid] = tmp1_y.x;
sdata_y1[tid] = tmp1_y.y;
sdata_y2[tid] = tmp1_y.z;
sdata_y3[tid] = tmp1_y.w;
__syncthreads();

float4 result_y;
result_y.x = a * sdata_x0[tid] + sdata_y0[tid];
result_y.y = a * sdata_x1[tid] + sdata_y1[tid];
result_y.z = a * sdata_x2[tid] + sdata_y2[tid];
result_y.w = a * sdata_x3[tid] + sdata_y3[tid];
y_as_float4[idx] = result_y;
}

}