#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_diff(float* a, const float* x, const float* y, const float c, int size){
int i = blockIdx.x * blockDim.x + threadIdx.x;
if( i < size )
a[i] += c*(x[i] - y[i]);
}