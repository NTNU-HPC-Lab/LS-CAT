#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float d_square_prox(float x0, float c, float f, float tau) {
return (x0 + 2.f * tau * c * f) / (1.f + 2.f * tau * c * c);
}
__device__ void d_calcDivergence(const float *v1, const float *v2, float &divv, size_t width, size_t height, size_t c, const bool *mask) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;
size_t i_mask = x + y * width;
size_t i      = x + y*width + c * width * height;

float v1x = 0.f, v2y = 0.f;
if (x>0 && mask[i_mask] && mask[i_mask-1]) v1x = v1[i] - v1[i-1];
if (y>0 && mask[i_mask] && mask[i_mask-width]) v2y = v2[i] - v2[i-width];
divv = -( v1x + v2y );
}
__global__ void g_updatePrimalVar(float *u, float *u_bar, float *u_diff, const float *p, const float *f, const float *scalar_op, float tau, float theta, size_t width, size_t height, size_t channels, const bool *mask) {
const int x = blockIdx.x * blockDim.x + threadIdx.x;
const int y = blockIdx.y * blockDim.y + threadIdx.y;

if(x>=width || y>=height) return;
if(!mask[x+y*width]) return;

for(int c = 0; c < channels; c++) {
const size_t i = x + y * width + c * width * height;
const float u_old = u[i];

float divp;
d_calcDivergence( &p[0], &p[width*height*channels], divp, width, height, c, mask );

const float u_new = d_square_prox(u_old - tau * divp, scalar_op[i], f[i], tau);
u_bar[i] = u_new + theta * (u_new - u_old);
u[i] = u_new;
u_diff[i] = abs(u_new - u_old);
}
}