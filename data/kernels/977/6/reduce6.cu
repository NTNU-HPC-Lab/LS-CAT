#include "hip/hip_runtime.h"
#include "includes.h"





template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
__global__ void reduce6(int *g_idata, int *g_odata, int n) {

extern __shared__ int sdata[];
// perform first level of reduction, reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
unsigned int gridSize = blockSize*2*gridDim.x;
sdata[tid] = 0;
while (i < n) {
sdata[tid] += g_idata[i] + g_idata[i+blockSize];
i += gridSize;
}
__syncthreads();

// do reduction in shared mem
if (blockSize >= 512) {
if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
}
if (blockSize >= 256) {
if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
}
if (blockSize >= 128) {
if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
}

if (tid < 32) warpReduce<blockSize>(sdata, tid);

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}