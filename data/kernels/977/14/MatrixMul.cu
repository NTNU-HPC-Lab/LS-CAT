#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
int accu = 0;

// Block index
int bx = blockIdx.x;
int by = blockIdx.y;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

int i = by * blockDim.y + ty;
int j = bx * blockDim.x + tx;

for(int k=0; k<width; k++)
{
accu = accu + M[i*width+k]*N[k*width+j];
}

P[i*width+j] = accu;
}