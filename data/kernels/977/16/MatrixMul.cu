#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
int bx = blockIdx.x;
int by = blockIdx.y;

int tx = threadIdx.x;
int ty = threadIdx.y;

//int i = by * blockDim.y + ty;
//int j = bx * blockDim.x + tx;

const int tile_size = 16;

__shared__ int As[tile_size][tile_size];
__shared__ int Bs[tile_size][tile_size];

int aBegin = width * tile_size * by;
int aEnd   = aBegin + width - 1;
int aStep  = tile_size;

int bBegin = tile_size * bx;
int bStep  = tile_size * width;

int Csub = 0;
int a, b;

for (a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
{
As[ty][tx] = M[a + width * ty + tx];
Bs[tx][ty] = N[b + width * tx + ty];  // Not memory coelescing !!!
__syncthreads();

for (int k = 0; k < tile_size; ++k)
{
Csub += As[ty][k] *  Bs[k][tx];  // No Bank Conflict
}
__syncthreads();
}

int c = width * tile_size * by + tile_size * bx;
P[c + width * ty + tx] = Csub;
}