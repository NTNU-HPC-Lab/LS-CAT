#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
int bx = blockIdx.x;
int by = blockIdx.y;

int tx = threadIdx.x;
int ty = threadIdx.y;

//int i = by * blockDim.y + ty;
//int j = bx * blockDim.x + tx;

const int tile_size = 16;

__shared__ int As[tile_size][tile_size];
__shared__ int Bs[tile_size][tile_size];

int aBegin = width * tile_size * by;
int aEnd   = aBegin + width - 1;
int aStep  = tile_size;

int bBegin = tile_size * bx;
int bStep  = tile_size * width;

int Csub = 0;
int a, b;

for (a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
{
As[ty][tx] = M[a + width * ty + tx]; // Memory coelescing !
Bs[tx][ty] = N[b + width * ty + tx]; // Memory coelescing !
__syncthreads();

for (int k = 0; k < tile_size; ++k)
{
// For Memory Coalescing :  Bs[k][tx] -> Bs[tx][k]
Csub += As[ty][k] *  Bs[tx][k]; // Bank Conflict on Bs[tx][k]
// It causes Bank Conflict
}
__syncthreads();
}

int c = width * tile_size * by + tile_size * bx;
P[c + width * ty + tx] = Csub;
}