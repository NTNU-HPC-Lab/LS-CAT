#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
int bx = blockIdx.x;
int by = blockIdx.y;

int tx = threadIdx.x;
int ty = threadIdx.y;

//int i = by * blockDim.y + ty;
//int j = bx * blockDim.x + tx;

const int tile_size = 16; // tile size

__shared__ int As[tile_size][tile_size];
__shared__ int Bs[tile_size][tile_size];

int aBegin = width * tile_size * by;
int aEnd   = aBegin + width - 1;
int aStep  = tile_size;

int bBegin = tile_size * bx;
int bStep  = tile_size * width;

int Csub = 0;
int a, b;

for (a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
{
As[ty][tx] = M[a + width * ty + tx];
Bs[ty][tx] = N[b + width * ty + tx];
__syncthreads();

//for (int k = 0; k < tile_size; ++k)
//{
//    Csub += As[ty][k] *  Bs[k][tx];
//}
// Loop Unrolling
Csub += As[ty][0] * Bs[0][tx];
Csub += As[ty][1] * Bs[1][tx];
Csub += As[ty][2] * Bs[2][tx];
Csub += As[ty][3] * Bs[3][tx];
Csub += As[ty][4] * Bs[4][tx];
Csub += As[ty][5] * Bs[5][tx];
Csub += As[ty][6] * Bs[6][tx];
Csub += As[ty][7] * Bs[7][tx];
Csub += As[ty][8] * Bs[8][tx];
Csub += As[ty][9] * Bs[9][tx];
Csub += As[ty][10] * Bs[10][tx];
Csub += As[ty][11] * Bs[11][tx];
Csub += As[ty][12] * Bs[12][tx];
Csub += As[ty][13] * Bs[13][tx];
Csub += As[ty][14] * Bs[14][tx];
Csub += As[ty][15] * Bs[15][tx];
__syncthreads();
}

int c = width * tile_size * by + tile_size * bx;
P[c + width * ty + tx] = Csub;
}