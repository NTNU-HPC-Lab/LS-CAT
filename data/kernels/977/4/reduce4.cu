#include "hip/hip_runtime.h"
#include "includes.h"





template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
__global__ void reduce4(int *g_idata, int *g_odata) {

extern __shared__ int sdata[];
// perform first level of reduction, reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}
if (tid < 32) warpReduce<512>(sdata, tid);

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}