#include "hip/hip_runtime.h"
#include "includes.h"


__device__ void showProgress(char* trystr, int tam, unsigned long long int id)
{
int n=0;

for(int i=0;i<tam;i++)
{
if('!' != trystr[i])
{
n=1;
break;
}
}

if(id % 1000000000 == 0) printf("\n----- Id = %lld\n", id);

if(n == 0) printf("\n***** %lld, Tam = %d -> ┤%s├\n", id, tam, trystr);
}
__device__ char * id2str(unsigned long long int n)
{
//Convert the id of the thread into the string in order to be checked
int b=96, r, asciChar, count=0;
char * a, * res;

a = (char*) malloc(100);

do
{
r=n%b;
asciChar=32+r;
a[count]=asciChar;
count++;
n=n/b;
}
while(n!=0);

res = (char*) malloc(count+1);

for(int i=count-1; i>=0; --i) res[count-i-1] = a[i];

free(a);

res[count] = '\0';

return res;
}
__device__ __host__  int shc(char* ca, int tam)
{
int h = 0;

for (int i = 0; i < tam; i++)
{
h = 31 * h + ca[i];
}

return h;
}
__global__ void findcollisions(int hash, unsigned long long int strLength)
{
unsigned long long int id = blockDim.x * blockIdx.x + threadIdx.x + strLength; //+ i* /*4e40*/;

char *trystr = id2str(id);
int hc;

int tam = 0;

while(trystr[tam]!='\0')
tam++;


//if (oldId < 50) printf("%lld, ┤%s├\n", id, trystr);
showProgress(trystr, tam, id);

hc = shc(trystr, tam);

if(hc == hash)
printf("Collision found for string ┤%s├. Hashcode %d\n", trystr, hc);

if(trystr[tam-1] == ' ')
{
char * reverse = (char *) malloc(tam+1);

for(int i=tam-1; i>=0; --i)
reverse[tam-i-1] = trystr[i];

reverse[tam] = '\0';

hc = shc(reverse, tam);

if(hc == hash)
printf("Collision found for string ┤%s├. Hashcode %d\n", reverse, hc);

free(reverse);
}
free(trystr);

}