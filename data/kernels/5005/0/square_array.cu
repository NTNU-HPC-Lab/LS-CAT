#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx<N) a[idx] = a[idx] * a[idx];
}