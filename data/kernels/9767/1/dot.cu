#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float*a, float*b, float*c, int threadperblock, int max){
__shared__ float cache[ThreadPerBlock];
int tid = threadIdx.x + blockDim.x*blockIdx.x;
float temp = 0;
int cacheindex = threadIdx.x;
while (tid < max){
temp = a[tid] * b[tid];
tid += gridDim.x*blockDim.x;
}
cache[cacheindex] = temp;

__syncthreads();
int i = blockDim.x / 2;
while (i != 0){
if (cacheindex < i)
cache[cacheindex] += cache[cacheindex + i];
__syncthreads();
i /= 2;
}
if (cacheindex == 0)
c[blockIdx.x] = cache[0];



}