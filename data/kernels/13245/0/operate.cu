#include "hip/hip_runtime.h"
#include "includes.h"

#define SEED
#define BLOCK_SIZE 32

typedef struct _data {
char * values;
char * next_values;
int width;
int height;
} data;

__global__ void operate(char * source, char * goal, int sizex, int sizey) {
__shared__ char local[BLOCK_SIZE + MASK_WIDTH - 1][BLOCK_SIZE + MASK_WIDTH - 1];
int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

int index = i * sizex + j;

int prim_x = j - MASK_RADIUS;
int first_x = prim_x;
for(; first_x - prim_x + threadIdx.x < MASK_WIDTH + BLOCK_SIZE - 1; first_x += BLOCK_SIZE) {
int prim_y = i - MASK_RADIUS;
int first_y = prim_y;
for(; first_y - prim_y + threadIdx.y < MASK_WIDTH + BLOCK_SIZE - 1; first_y += BLOCK_SIZE) {
if(first_y >= 0 && first_y < sizey && first_x >= 0 && first_x < sizex) {
local[first_y - prim_y + threadIdx.y][first_x - prim_x + threadIdx.x] =
source[first_y * sizex + first_x];
}
else {
local[first_y - prim_y + threadIdx.y][first_x - prim_x + threadIdx.x] = '0';
}
}
}
__syncthreads();

if(i < sizey && j < sizex) {
int l_j, l_i;
int amount = 0;

for(l_i = 0; l_i < MASK_WIDTH; l_i++) {
if( ( (int) threadIdx.y + l_i >= 0 ) && ( (int) threadIdx.y + l_i < BLOCK_SIZE + MASK_WIDTH - 1) ) {
for(l_j = 0; l_j < MASK_WIDTH; l_j++){
if( ( (int) threadIdx.x + l_j >= 0 ) && ( (int) threadIdx.x + l_j < BLOCK_SIZE + MASK_WIDTH - 1) ) {
if(local[threadIdx.y + l_i][threadIdx.x + l_j] == '1')
amount++;
}
}
}
}

if(source[index] == '1')
amount--;

if(source[index] == '1') {
if(amount < 2 || amount > 3)
goal[index] = '0';
else
goal[index] = '1';
}
else {
if(amount == 3)
goal[index] = '1';
else
goal[index] = '0';
}
}
}