#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce0(float *g_idata, float *g_odata, int N){
extern __shared__ float sdata[];
// each thread loads one element from global to shared mem
int tid = threadIdx.x;
int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
if((i+blockDim.x)<N)
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
else if(i<N)
sdata[tid] = g_idata[i];
else
sdata[tid] = 0.0;
__syncthreads();
// do reduction in shared mem
for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
if (tid < s) {
sdata[tid] += sdata[tid + s];
}
__syncthreads();
}
if (tid < 32)
{
sdata[tid] += sdata[tid + 32];
sdata[tid] += sdata[tid + 16];
sdata[tid] += sdata[tid + 8];
sdata[tid] += sdata[tid + 4];
sdata[tid] += sdata[tid + 2];
sdata[tid] += sdata[tid + 1];
}

// do reduction in shared mem

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}