#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rowDiv(float* a, float* b, float* c, int M, int N){

int i = blockIdx.x*blockDim.x + threadIdx.x;
c[i] = a[i]/b[blockIdx.x];
}