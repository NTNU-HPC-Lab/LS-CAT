#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colMul(float* a, float* b, float* c, int M, int N){

int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<M){
int ind = i + blockIdx.y*M;
c[ind] = a[ind]*b[i];
}
}