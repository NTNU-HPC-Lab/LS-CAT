#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* a, float* b, float* c, const int N)
{
const int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<N)
c[i] = a[i] + b[i];
}