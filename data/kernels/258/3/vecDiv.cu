#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecDiv(float* a,float* b,float* c,const int N){
const int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<N)
c[i] = __fdividef(a[i],b[i]);
//c[i] = a[i]/b[i];
}