#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecEps(float* a,const int N){
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(a[i] < EPS && i < N)
a[i] = EPS;
}