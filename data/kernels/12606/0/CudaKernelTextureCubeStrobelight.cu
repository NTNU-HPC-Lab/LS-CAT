#include "hip/hip_runtime.h"
#include "includes.h"
/* ==========================================================================
textureCube.cu
==========================================================================

Main wrapper + kernel that changes the colors of the four faces

*/




#define PI 3.1415926536f

// --------------------------------------------------------------------------
// Kernel
// --------------------------------------------------------------------------


// Paint a 2D surface with a moving bulls-eye pattern.  The "face" parameter selects
// between 6 different colors to use.  We will use a different color on each face of a
// cube map.


// --------------------------------------------------------------------------
// Wrapper
// --------------------------------------------------------------------------

// Sets up grid / blocks, launches kernel
extern "C"
__global__ void CudaKernelTextureCubeStrobelight(char *surface, int width, int height, size_t pitch, int face, float t)
{
int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;
unsigned char *pixel;

// in the case where, due to quantization into grids, we have
// more threads than pixels, skip the threads which don't
// correspond to valid pixels
if (x >= width || y >= height) return;

// get a pointer to this pixel
pixel = (unsigned char *)(surface + y*pitch) + 4 * x;

// populate it
float theta_x = (2.0f*x) / width - 1.0f;
float theta_y = (2.0f*y) / height - 1.0f;
float theta = 2.0f*PI*sqrt(theta_x*theta_x + theta_y*theta_y);
unsigned char value = 255 * (0.6f + 0.4f*cos(theta + t));

pixel[3] = 255; // alpha

if (face % 2)
{
pixel[0] =    // blue
pixel[1] =    // green
pixel[2] = 0.5; // red
pixel[face / 2] = value;
}
else
{
pixel[0] =        // blue
pixel[1] =        // green
pixel[2] = value; // red
pixel[face / 2] = 0.5;
}
}