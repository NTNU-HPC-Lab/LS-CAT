#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( double *a, double *b, double *c, int n )
{
int tid = threadIdx.x + blockIdx.x * blockDim.x;
// handle the data at this index
while (tid < n) {
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}
//printf("Value of *ip variable: %f\n", a[tid] );

}