#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSumTable(const float *rowCumSum, float *SumTable, int rowNumberN, int colNumberM) {
int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
if (xIndex >= colNumberM) return;
for (int i = 1; i < rowNumberN; i++) {
SumTable[i * colNumberM + xIndex] +=
rowCumSum[(i - 1) * colNumberM + xIndex];
}
}