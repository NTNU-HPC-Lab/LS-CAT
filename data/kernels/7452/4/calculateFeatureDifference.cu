#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float computeS(float *sumTable, int rowNumberN, int colNumberM, int startX, int startY, int Kx, int Ky) {
startX--;
startY--;
float S =
sumTable[startX + Kx + (Ky + startY) * colNumberM] -
(startX < 0 ? 0 : sumTable[startX + (Ky + startY) * colNumberM]) -
(startY < 0 ? 0 : sumTable[startX + Kx + startY * colNumberM]) +
(startX < 0 || startY < 0 ? 0 : sumTable[startX + startY * colNumberM]);
return S;
}
__global__ void calculateFeatureDifference(float *templateFeatures, int colNumberM, int rowNumberN, float *l1SumTable, float *l2SumTable, float *lxSumTable, float *lySumTable, int Kx, int Ky, float *differences) {
int widthLimit = colNumberM - Kx + 1;
int heightLimit = rowNumberN - Ky + 1;

float meanVector;
float varianceVector;
float xGradientVector;
float yGradientVector;
int startX = threadIdx.x + blockIdx.x * blockDim.x;
int startY = threadIdx.y + blockIdx.y * blockDim.y;
if (startX >= widthLimit || startY >= heightLimit) return;
float S1D =
computeS(l1SumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);
float S2D =
computeS(l2SumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);

meanVector = S1D / (Kx * Ky);

varianceVector = S2D / (Kx * Ky) - powf(meanVector, 2);

float SxD =
computeS(lxSumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);

xGradientVector = 4 * (SxD - (startX + Kx / 2.0) * S1D) / (Kx * Kx * Ky);

float SyD =
computeS(lySumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);
yGradientVector = 4 * (SyD - (startY + Ky / 2.0) * S1D) / (Ky * Ky * Kx);

differences[startX + startY * widthLimit] = norm4df(
templateFeatures[0] - meanVector, templateFeatures[1] - varianceVector,
templateFeatures[2] - xGradientVector,
templateFeatures[3] - yGradientVector);
}