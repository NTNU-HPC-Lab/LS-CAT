#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2grayKernel(unsigned char *imgr,unsigned char *imgg,unsigned char *imgb,unsigned char *img_gray, int n) {

int r, g, b;
int index = threadIdx.x + blockIdx.x * blockDim.x;

if (index < n){
r = imgr[index];
g = imgg[index];
b = imgb[index];

img_gray[index] = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
}
}