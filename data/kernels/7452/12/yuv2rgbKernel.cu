#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yuv2rgbKernel(int *imgy,int *imgcb,int *imgcr, int *imgr,int *imgg,int *imgb, int n) {

int r, g, b;
int y, cb, cr;

int index;
index = threadIdx.x + blockIdx.x * blockDim.x;

if (index < n){
y = imgy[index];
cb = imgcb[index];
cr = imgcr[index];

r  = (int)( 1*y + 0*cb +  1.14*cr);
g = (int)( 1*y - 0.396*cb -  0.581*cr);
b = (int)( 1*y + 2.029*cb +  0*cr);

imgr[index] = r;
imgg[index] = g;
imgb[index] = b;
}
}