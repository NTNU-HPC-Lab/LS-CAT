#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2yuvKernel(int *imgr,int *imgg,int *imgb,int *imgy,int *imgcb,int *imgcr, int n) {

int r, g, b;
int y, cb, cr;

int index;
index = threadIdx.x + blockIdx.x * blockDim.x;

if (index < n){
r = imgr[index];
g = imgg[index];
b = imgb[index];

y  = (int)( 0.299*r + 0.587*g +  0.114*b);
cb = (int)(-0.147*r - 0.289*g +  0.436*b);
cr = (int)( 0.615*r - 0.515*g - 0.1*b);

imgy[index] = y;
imgcb[index] = cb;
imgcr[index] = cr;
}
}