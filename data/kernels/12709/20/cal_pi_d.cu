#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cal_pi_d(double *sum, int nbin, double step, int nthreads, int nBLOCKS) {
int i;
double x;
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the BLOCKS
for (i=idx; i< nbin; i+=nthreads*nBLOCKS) {
x = (i+0.5)*step;
sum[idx] += 4.0/(1.0+x*x);
}
}