#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dotProduct_CUDA_float(float *sum, int size, float *vector1, float *vector2){
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
if(idx < size){
sum[idx] = (vector2[idx]) * (vector1[idx]);
}
}