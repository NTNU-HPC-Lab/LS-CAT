#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cal_pi_d(double *sum, int nbin, double step, int nthreads, int nblocks) {
int i;
double x;
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
for (i=idx; i< nbin; i+=nthreads*nblocks) {
x = (i+0.5)*step;
sum[idx] += 4.0/(1.0+x*x);
}
}