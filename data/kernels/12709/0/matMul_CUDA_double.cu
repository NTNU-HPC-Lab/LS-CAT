#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREAD  256  // Number of thread blocks
#define print(x) printf("%d",x)

float *matrixMul_float_serial(float vector1[], float vector2[], int size);
float *matrixMul_float_parallel(float vector1[], float vector2[], int size, int thread_count);
float *matrixMul_float_cuda(float* vector1, float* vector2, int num);
double *matrixMul_double_serial(double vector1[], double vector2[], int size);
double *matrixMul_double_parallel(double vector1[], double vector2[], int size, int thread_count);
double *matrixMul_double_cuda(double* vector1, double* vector2, int num);
double doubleGen();
float floatGen();
void operations(int size, int parallel, int serial, int cuda, int verify, int thread_count);
void print_results_float( int size, double time_spent);
void print_results_double( int size, double time_spent);
double verifyVectord(double *vector1, double *vector2, int size);
float verifyVectorf(float *vector1, float *vector2, int size);



__global__ void matMul_CUDA_double(double *sum, int size, double *vector1, double *vector2){
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
int k;
if(idx < size*size){
for(k=0; k< size; k++){
sum[idx] += (*(vector1+(idx-(idx % size)+k))) * (*(vector2+(k*size+(idx % size))));
}
}
}