#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dotProduct_CUDA(double *sum, long size, double *vector1, double *vector2){
long idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
if(idx < size){
//printf("Before idx%d : %lf\n",idx,sum[idx]);
sum[idx] = (vector2[idx]) * (vector1[idx]);
//printf("Vector1 %lf\n",*(vector1+idx));
//printf("Vector2 %lf\n",vector2[idx]);
//printf("After idx%d : %lf\n",idx,sum[idx]);
}
}