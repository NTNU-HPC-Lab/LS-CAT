#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dotProduct_CUDA_double(double *sum, int size, double *vector1, double *vector2){
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
if(idx < size){
sum[idx] = (vector2[idx]) * (vector1[idx]);
}
}