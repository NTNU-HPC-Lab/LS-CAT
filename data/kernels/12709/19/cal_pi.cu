#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nBLOCKS) {
int i;
float x;
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the BLOCKS
for (i=idx; i< nbin; i+=nthreads*nBLOCKS) {
x = (i+0.5)*step;
sum[idx] += 4.0/(1.0+x*x);
}
}