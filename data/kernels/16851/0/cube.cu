#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;


__global__ void cube(long *deviceOutput, long *deviceInput)
{
int idx = threadIdx.x;
long f = deviceInput[idx];
deviceOutput[idx] = f * f * f;
}