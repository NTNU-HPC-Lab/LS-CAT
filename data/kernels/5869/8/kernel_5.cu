#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_5(float *d_data_in, float *d_data_out, int data_size)
{
__shared__ volatile float s_data[BLKSIZE];
int tid = threadIdx.x;
int index = tid + blockIdx.x*blockDim.x*2;
s_data[tid] = 0.0;
if (index < data_size){
s_data[tid] = d_data_in[index];
}
if ((index + blockDim.x) < data_size){
s_data[tid] += d_data_in[index + blockDim.x];
}
__syncthreads();

for (int s = blockDim.x / 2; s >= 64; s = s >> 1){
if (tid<s){
s_data[tid] += s_data[tid + s];
}
__syncthreads();
}

if (tid < 32){
s_data[tid] += s_data[tid + 32];
s_data[tid] += s_data[tid + 16];
s_data[tid] += s_data[tid + 8];
s_data[tid] += s_data[tid + 4];
s_data[tid] += s_data[tid + 2];
s_data[tid] += s_data[tid + 1];
}

if (tid == 0){
d_data_out[blockIdx.x] = s_data[tid];
}
}