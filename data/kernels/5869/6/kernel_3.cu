#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_3(float *d_data_in, float *d_data_out, int data_size)
{
__shared__ float s_data[BLKSIZE];
int tid = threadIdx.x;
int index = tid + blockIdx.x*blockDim.x;
s_data[tid] = 0.0;
if (index < data_size){
s_data[tid] = d_data_in[index];
}
__syncthreads();

for (int s = blockDim.x/2; s >= 1; s = s >> 1){
if (tid<s){
s_data[tid] += s_data[tid + s];
}
__syncthreads();
}

if (tid == 0){
d_data_out[blockIdx.x] = s_data[tid];
}
}