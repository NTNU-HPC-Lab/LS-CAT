#include "hip/hip_runtime.h"
#include "includes.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))
__global__ void pos_update(int nx, int ny, double dt, double* d_z, double* d_v, double* d_a) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
int r = i / nx;
int c = i % nx;
if(r<ny-1 && r>0 && c<nx-1 && c>0){
d_v[i] = d_v[i] + dt*d_a[i];
d_z[i] = d_z[i] + dt*d_v[i];
}
}