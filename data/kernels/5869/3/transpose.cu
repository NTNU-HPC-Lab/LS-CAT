#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose(double *in_d, double * out_d, int row, int col)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
out_d[y+col*x] = in_d[x+row*y];
}