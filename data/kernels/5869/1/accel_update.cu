#include "hip/hip_runtime.h"
#include "includes.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))
__global__ void accel_update(int nx, int ny, double dx2inv, double dy2inv, double* d_z, double* d_a) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
double ax, ay;
int r = i / nx;
int c = i % nx;
if(i < nx*ny) {
if(r<ny-1 && r>0 && c<nx-1 && c>0){
ax = (d_z[i+nx]+d_z[i-nx]-2.0*d_z[i])*dx2inv;
ay = (d_z[i+1]+d_z[i-1]-2.0*d_z[i])*dy2inv;
d_a[i] = (ax+ay)/2;
}
else
d_a[i] = 0.0;
}
}