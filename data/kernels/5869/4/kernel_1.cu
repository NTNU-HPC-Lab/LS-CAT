#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_1(float *d_data_in, float *d_data_out, int data_size)
{
__shared__ float s_data[BLKSIZE];
int tid = threadIdx.x;
int index = tid + blockIdx.x*blockDim.x;
s_data[tid] = 0.0;
if (index < data_size){
s_data[tid] = d_data_in[index];
}
__syncthreads();

for (int s = 2; s <= blockDim.x; s = s * 2){
if ((tid%s) == 0){
s_data[tid] += s_data[tid + s / 2];
}
__syncthreads();
}

if (tid == 0){
d_data_out[blockIdx.x] = s_data[tid];
}
}