#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_VALUE 10


__global__ void saxpy(float *X, float *Y, float *Z, int A, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if(i<N){
Z[i] = A * X[i] + Y[i];
}
}