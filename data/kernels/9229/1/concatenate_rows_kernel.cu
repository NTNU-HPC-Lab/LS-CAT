#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int concatenate(float* array)
{
unsigned int rvalue=0;
unsigned int sign;

for (int i = 0; i < 32; i++)
{
sign = (array[i]>=0);
rvalue = rvalue | (sign<<i);
}

return rvalue;
}
__global__ void concatenate_rows_kernel(float *a, unsigned int *b, int size)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i<size) b[i] = concatenate(&a[i*32]);
}