#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xnor_gemm(unsigned int* A, unsigned int* B, float* C, int m, int n, int k) {

// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

// Thread row and column within Csub
int row = threadIdx.y;
int col = threadIdx.x;

// Each thread block computes one sub-matrix Csub of C
float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

// Shared memory used to store Asub and Bsub respectively
__shared__ unsigned int As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ unsigned int Bs[BLOCK_SIZE][BLOCK_SIZE];

// Each thread computes one element of Csub
// by accumulating results into Cvalue
// block_size = 16 -> 256 threads, one per Csub element
unsigned int Cvalue = 0;

// Loop over all the sub-matrices of A and B that are
// required to compute Csub
// Multiply each pair of sub-matrices together
// and accumulate the results
for (int i = 0; i < (n / BLOCK_SIZE); ++i) {

// Get sub-matrix Asub of A
unsigned int* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];

// Get sub-matrix Bsub of B
unsigned int* Bsub = &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];

// Load Asub and Bsub from device memory to shared memory
// Each thread loads one element of each sub-matrix
As[row][col] = Asub[row*n+col];
Bs[row][col] = Bsub[row*k+col];

// Synchronize to make sure the sub-matrices are loaded
// before starting the computation
__syncthreads();

// Multiply Asub and Bsub together
// THIS IS THE MOST INTERESTING PART
for (int j = 0; j < BLOCK_SIZE; ++j) Cvalue += __popc(As[row][j]^Bs[j][col]);

// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write Csub to device memory
// Each thread writes one element
if(col + blockCol* BLOCK_SIZE< k && row + blockRow* BLOCK_SIZE< m) Csub[row*k+col] = -(2*(float)Cvalue-32*n);
}