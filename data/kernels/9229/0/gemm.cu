#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gemm(float* A, float* B, float* C, int m, int n, int k) {

// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;

// Thread row and column within Csub
int row = threadIdx.y;
int col = threadIdx.x;

// Each thread block computes one sub-matrix Csub of C
float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

// Shared memory used to store Asub and Bsub respectively
__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

// Each thread computes one element of Csub
// by accumulating results into Cvalue
// block_size = 16 -> 256 threads, one per Csub element
float Cvalue = 0.0;

// Loop over all the sub-matrices of A and B that are
// required to compute Csub
// Multiply each pair of sub-matrices together
// and accumulate the results
for (int i = 0; i < (n / BLOCK_SIZE); ++i) {

// Get sub-matrix Asub of A
float* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];

// Get sub-matrix Bsub of B
float* Bsub = &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];

// Load Asub and Bsub from device memory to shared memory
// Each thread loads one element of each sub-matrix
As[row][col] = Asub[row*n+col];
Bs[row][col] = Bsub[row*k+col];

// Synchronize to make sure the sub-matrices are loaded
// before starting the computation
__syncthreads();

// Multiply Asub and Bsub together
for (int j = 0; j < BLOCK_SIZE; ++j) Cvalue += As[row][j] * Bs[j][col];

// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write Csub to device memory
// Each thread writes one element
if(col + blockCol* BLOCK_SIZE< k && row + blockRow* BLOCK_SIZE< m) Csub[row*k+col] = Cvalue;
}