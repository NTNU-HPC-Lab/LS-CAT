#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float* deconcatenate(unsigned int x)
{
float * array = new float[32];

for (int i = 0; i < 32; i++)
{
array[i] = (x & ( 1 << i )) >> i;
}

return array;
}
__device__ unsigned int concatenate(float* array)
{
unsigned int rvalue=0;
unsigned int sign;

for (int i = 0; i < 32; i++)
{
sign = (array[i]>=0);
rvalue = rvalue | (sign<<i);
}

return rvalue;
}
__global__ void deconcatenate_rows_kernel(unsigned int *a, float *b, int size)
{
float * array;

for(int i=0; i<size; i+=32)
{
array = deconcatenate(a[i/32]);
for (int k=0;k<32;k++) b[i+k] = array[k];
delete[] array;
}
}