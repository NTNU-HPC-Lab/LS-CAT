#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int concatenate(float* array)
{
unsigned int rvalue=0;
unsigned int sign;

for (int i = 0; i < 32; i++)
{
sign = (array[i]>=0);
rvalue = rvalue | (sign<<i);
}

return rvalue;
}
__global__ void concatenate_cols_kernel(float *a, unsigned int *b, int m, int n)
{

int j = blockIdx.x * blockDim.x + threadIdx.x;

if(j<n){
float * array = new float[32];
for(int i=0; i<m; i+=32){
for(int k=0; k<32;k++) array[k] = a[j + n*(i+k)];
b[j+n*i/32]=concatenate(array);
}
delete[] array;
}
}