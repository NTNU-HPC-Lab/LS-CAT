#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void vecInit(float* X, unsigned int size, float value)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < size; i += stride) {
X[i] = value;
}
}