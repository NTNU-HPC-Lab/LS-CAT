#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
extern __shared__ float sdata[];

// perform first level of reduction,
// reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
unsigned int gridSize = blockDim.x*2*gridDim.x;

float mySum = 0;

// we reduce multiple elements per thread.  The number is determined by the
// number of active thread blocks (via gridDim).  More blocks will result
// in a larger gridSize and therefore fewer elements per thread
while (i < n)
{
mySum += g_idata[i];
// ensure we don't read out of bounds
if (i + blockDim.x < n)
mySum += g_idata[i+blockDim.x];
i += gridSize;
}

// each thread puts its local sum into shared memory
sdata[tid] = mySum;
__syncthreads();


// do reduction in shared mem
if (blockDim.x >= 1024) { if (tid < 512) { sdata[tid] = mySum = mySum + sdata[tid + 512]; } __syncthreads(); }
if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

// avoid bank conflict
if (tid < 32)
{
// now that we are using warp-synchronous programming (below)
// we need to declare our shared memory volatile so that the compiler
// doesn't reorder stores to it and induce incorrect behavior.
volatile float* smem = sdata;
if (blockDim.x >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
if (blockDim.x >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
if (blockDim.x >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
if (blockDim.x >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
if (blockDim.x >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
if (blockDim.x >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
}

// write result for this block to global mem
if (tid == 0)
g_odata[blockIdx.x] = sdata[0];
}