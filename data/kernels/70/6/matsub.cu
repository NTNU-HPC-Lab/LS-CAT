#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void matsub( float* X, float* Y, unsigned int size)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned int i = idx; i < size; i += stride) {
X[i] -= Y[i];
}
}