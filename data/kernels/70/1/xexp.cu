#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/












__global__ void xexp( float* X, float* C, float* Y, float* Z, unsigned int size)
{
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
const unsigned int stride = blockDim.x * gridDim.x;

for (unsigned long int i = idx; i < size; i += stride) {
X[i] = Z[i]*__expf(C[i] - Y[i]);
}
}