#include "hip/hip_runtime.h"
#include "includes.h"
//using namespace Eigen;
using namespace std;

__device__ void setPhysicialParameters(float T, float *ce, float *pho, float *lamda)
{
float Ts = 1456.16f, Tl = 1522.69f, fs = 0.0f, L = 268000.0f;
if (T < Ts)
{
fs = 0;
*pho = 7250.0f;
*lamda = 50.0f;
*ce = 540.0f;
}

if (T >= Ts && T <= Tl)
{
fs = (Tl - T) / (Tl - Ts);
*pho = 7250.0f;
*lamda = fs * 25.0f + (1.0f - fs) * 50.0f;
*ce = 540.0f + L / (Tl - Ts);
}

if (T > Tl)
{
fs = 1;
*pho = 7250.0f;
*lamda = 28.0f;
*ce = 540.0f;
}
}
__device__ float setBoundaryCondition(int tstep, float tau, float Vcast, float *hPop, int Section, float *ccml)
{
float zposition = tstep * tau * fabs(Vcast);//ËÙ¶È³ËÒÔÊ±¼ä(Ê±¼äÍø¸ñ*Íø¸ñÊý£©,¸÷¸öÀäÈ´¶Î³¤¶È
float h = 0; //±íÃæ´«ÈÈÏµÊý

for (int i = 0; i < Section; i++)
{
if (zposition >= *(ccml + i) && zposition <= *(ccml + i + 1))//ÏÞ¶¨¸÷¸öÀäÈ´¶Î£¬Ã¿¸öÀäÈ´¶Î¶ÔÓ¦Ò»¸öh
{
h = *(hPop + blockIdx.x * Section + i);
}
}
return h;
}
__global__ void solvePDEKernel(float *hPop, float *T_Last, float *T_New, float *T_Surface, float Tw, float lamda, float pho, float ce, int ny, float dy, int nx, float dx, float tau, int tnpts, int tstep, float Vcast, int Section, float *ccml)
{
float ax, ay, T_Up, T_Down, T_Middle, T_Right, T_Left;
float h;
ax = tau * lamda / (pho * ce * dx * dx);
ay = tau * lamda / (pho * ce * dy * dy);

int i = threadIdx.x;
int j = threadIdx.y;
int tis = blockIdx.x * nx * ny + i * ny + j;
int L = ny;

setPhysicialParameters(T_Last[tis], &ce, &pho, &lamda);
h = setBoundaryCondition(tstep, tau, Vcast, hPop, Section, ccml);

if (i != 0 && i != (nx - 1) && j != 0 && j != (ny - 1))//ÖÐ¼ä
{
T_Right = T_Last[tis + L];
T_Left = T_Last[tis - L];
T_Middle = T_Last[tis];
T_Up = T_Last[tis + 1];
T_Down = T_Last[tis - 1];
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == 0 && j == 0)//µã1
{
T_Up = T_Last[tis + 1];
T_Middle = T_Last[tis];
T_Down = T_Last[tis + 1] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_Right = T_Last[tis + L];
T_Left = T_Last[tis + L] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == (nx - 1) && j == 0)//µã2
{
T_Up = T_Last[tis + 1];
T_Middle = T_Last[tis];
T_Down = T_Last[tis + 1] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_Left = T_Last[tis - L];
T_Right = T_Last[tis - L] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == 0 && j == (ny - 1))//µã3
{
T_Up = T_Last[tis - 1] - 2 * dx *h * (T_Last[tis] - Tw) / lamda;
T_Middle = T_Last[tis];
T_Down = T_Last[tis - 1];
T_Right = T_Last[tis + L];
T_Left = T_Last[tis + L] - 2 * dx *h * (T_Last[tis] - Tw) / lamda;
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == (nx - 1) && j == (ny - 1))//µã4
{
T_Up = T_Last[tis - 1] - 2 * dx *h * (T_Last[tis] - Tw) / lamda;
T_Middle = T_Last[tis];
T_Down = T_Last[tis - 1];
T_Right = T_Last[tis - L] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_Left = T_Last[tis - L];
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == 0 && j != 0 && j != (ny - 1))//±ß1
{
T_Up = T_Last[tis + 1];
T_Middle = T_Last[tis];
T_Down = T_Last[tis - 1];
T_Right = T_Last[tis + L];
T_Left = T_Last[tis + L] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i == (nx - 1) && j != 0 && j != (ny - 1))//±ß2
{
T_Up = T_Last[tis + 1];
T_Middle = T_Last[tis];
T_Down = T_Last[tis - 1];
T_Right = T_Last[tis - L] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_Left = T_Last[tis - L];
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i != 0 && i != (nx - 1) && j == 0)//±ß3
{
T_Up = T_Last[tis + 1];
T_Middle = T_Last[tis];
T_Down = T_Last[tis + 1] - 2 * dx * h* (T_Last[tis] - Tw) / lamda;
T_Right = T_Last[tis + L];
T_Left = T_Last[tis - L];
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
else if (i != 0 && i != (nx - 1) && j == (ny - 1))//±ß4
{
T_Up = T_Last[tis - 1] - 2 * dx * h * (T_Last[tis] - Tw) / lamda;
T_Middle = T_Last[tis];
T_Down = T_Last[tis - 1];
T_Right = T_Last[tis + L];
T_Left = T_Last[tis - L];
T_New[tis] = ax * T_Right - (2 * ax + 2 * ay - 1) * T_Middle + ax * T_Left + ay * T_Up + ay * T_Down;
}
if (i == 0 && j == int((ny - 1)/2))
T_Surface[blockIdx.x * tnpts + tstep] = T_New[tis];
T_Last[tis] = T_New[tis];
__syncthreads();
}