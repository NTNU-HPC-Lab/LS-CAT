#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "solvePDEKernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *hPop = NULL;
hipMalloc(&hPop, XSIZE*YSIZE);
float *T_Last = NULL;
hipMalloc(&T_Last, XSIZE*YSIZE);
float *T_New = NULL;
hipMalloc(&T_New, XSIZE*YSIZE);
float *T_Surface = NULL;
hipMalloc(&T_Surface, XSIZE*YSIZE);
float Tw = 1;
float lamda = 1;
float pho = 1;
float ce = 1;
int ny = 1;
float dy = 1;
int nx = 1;
float dx = 1;
float tau = 1;
int tnpts = 1;
int tstep = 1;
float Vcast = 1;
int Section = 1;
float *ccml = NULL;
hipMalloc(&ccml, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
solvePDEKernel<<<gridBlock,threadBlock>>>(hPop,T_Last,T_New,T_Surface,Tw,lamda,pho,ce,ny,dy,nx,dx,tau,tnpts,tstep,Vcast,Section,ccml);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
solvePDEKernel<<<gridBlock,threadBlock>>>(hPop,T_Last,T_New,T_Surface,Tw,lamda,pho,ce,ny,dy,nx,dx,tau,tnpts,tstep,Vcast,Section,ccml);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
solvePDEKernel<<<gridBlock,threadBlock>>>(hPop,T_Last,T_New,T_Surface,Tw,lamda,pho,ce,ny,dy,nx,dx,tau,tnpts,tstep,Vcast,Section,ccml);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}