#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nms_kernel( const int num_per_thread, const float threshold, const int num_detections, const int *indices, float *scores, const float *classes, const float4 *boxes) {

// Go through detections by descending score
for (int m = 0; m < num_detections; m++) {
for (int n = 0; n < num_per_thread; n++) {
int i = threadIdx.x * num_per_thread + n;
if (i < num_detections && m < i && scores[m] > 0.0f) {
int idx = indices[i];
int max_idx = indices[m];
int icls = classes[idx];
int mcls = classes[max_idx];
if (mcls == icls) {
float4 ibox = boxes[idx];
float4 mbox = boxes[max_idx];
float x1 = max(ibox.x, mbox.x);
float y1 = max(ibox.y, mbox.y);
float x2 = min(ibox.z, mbox.z);
float y2 = min(ibox.w, mbox.w);
float w = max(0.0f, x2 - x1 + 1);
float h = max(0.0f, y2 - y1 + 1);
float iarea = (ibox.z - ibox.x + 1) * (ibox.w - ibox.y + 1);
float marea = (mbox.z - mbox.x + 1) * (mbox.w - mbox.y + 1);
float inter = w * h;
float overlap = inter / (iarea + marea - inter);
if (overlap > threshold) {
scores[i] = 0.0f;
}
}
}
}

// Sync discarded detections
__syncthreads();
}
}