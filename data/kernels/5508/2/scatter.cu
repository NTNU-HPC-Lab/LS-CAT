#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter(unsigned int *d_in, unsigned int *d_index, unsigned int *d_out, unsigned int in_size) {
unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
if (index < in_size) {
d_out[d_index[index]] = d_in[index];
}
}