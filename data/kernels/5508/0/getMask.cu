#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getMask(unsigned int *d_in, unsigned int *d_out, unsigned int in_size, unsigned int bit_shift, unsigned int One) {
unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
unsigned int bit = 0;
if (index < in_size) {
bit = d_in[index] & (1 << bit_shift);
bit = (bit > 0) ? 1 : 0;
d_out[index] = (One ? bit : 1 - bit);
}
}