#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getIndex(unsigned int *d_index, unsigned int *d_scan, unsigned int *d_mask, unsigned int in_size, unsigned int total_pre) {
unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

if (index < in_size) {
if (d_mask[index] == 1) {
d_index[index] = total_pre + d_scan[index];
}
}
}