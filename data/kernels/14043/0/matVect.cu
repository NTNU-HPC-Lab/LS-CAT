#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Rayhana ZIARA
* produit matrice vecteur
*/


/*
* DESCRIPTION : kernel concernant le produit matrice vecteur
* PARAMETRES : matrice A, vecteur v, vecteur r et taille des vecteurs
* RETOUR : /
*/

/*
* DESCRIPTION : fonction d'affichage de matrice et de vecteur
* PARAMETRES : matrice à afficher, nb ligne et nb colonne de A,
* RETOUR : /
*/
__global__ void matVect(float *A, float *v, float *r, int size)
{
float resultat = 0.0;
int index = blockIdx.x * blockDim.x + threadIdx.x;

if(index > size)
{
printf("ERREUR - Index > size\n");
return;
}

for(int i = 0; i < size; i++)
resultat += A[i * size + index] * v[i];

r[index] = resultat;
}