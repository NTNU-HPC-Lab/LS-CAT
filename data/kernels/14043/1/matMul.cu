#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(float *A, int l_A, int c_A, float *B, int l_B, int c_B, float *C, int l_C, int c_C)
{
float resultat = 0.0;
int ligne = blockDim.x * blockIdx.x + threadIdx.x;
int colonne = blockDim.y * blockIdx.y + threadIdx.y;

if(ligne > l_A || colonne > c_B)
{
printf("ERREUR - Soit ligne > m soit colonne > m\n");
return;
}

for(int i = 0; i < c_A; i++)
resultat += A[ligne * c_A + i] * B[i * c_B + colonne];

C[ligne * c_C + colonne] = resultat;
}