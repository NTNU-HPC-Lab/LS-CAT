#include "hip/hip_runtime.h"
#include "includes.h"

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;


#define SHARED_MEMORY_SIZE 49152
#define MERGE_THREADBLOCK_SIZE 128

static uint *d_PartialHistograms;

/*
*	Function that maps value to bin in range 0 inclusive to binCOunt exclusive
*/
__global__ void mergePartialHistogramsKernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount,	uint binCount)
{
for (uint bin = blockIdx.x; bin < binCount; bin += gridDim.x)
{
uint sum = 0;
for (uint histogramIndex = threadIdx.x; histogramIndex < histogramCount; histogramIndex += MERGE_THREADBLOCK_SIZE)
{
sum += d_PartialHistograms[bin + histogramIndex * binCount];
}

__shared__ uint data[MERGE_THREADBLOCK_SIZE];
data[threadIdx.x] = sum;

for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
{
__syncthreads();

if (threadIdx.x < stride)
{
data[threadIdx.x] += data[threadIdx.x + stride];
}
}

if (threadIdx.x == 0)
{
d_Histogram[bin] = data[0];
}
}
}