#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeHistogram256Kernel( uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount )
{
uint sum = 0;

//for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)//MERGE_THREADBLOCK_SIZE->HISTOGRAM256_BIN_COUNT ??
for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)//original
{
//sum += d_PartialHistograms[blockIdx.x + i * MERGE_THREADBLOCK_SIZE];
sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];//original
}

//__shared__ uint data[HISTOGRAM256_THREADBLOCK_SIZE];
__shared__ uint data[MERGE_THREADBLOCK_SIZE];//original
data[threadIdx.x] = sum;

for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
{
__syncthreads();

//if (threadIdx.x < stride && threadIdx.x + stride < HISTOGRAM256_THREADBLOCK_SIZE)
if (threadIdx.x < stride)//original
{
data[threadIdx.x] += data[threadIdx.x + stride];
}
}

if (threadIdx.x == 0)
{
d_Histogram[blockIdx.x] = data[0];
}
}