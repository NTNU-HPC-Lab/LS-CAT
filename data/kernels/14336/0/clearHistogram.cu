#include "hip/hip_runtime.h"
#include "includes.h"

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;


#define SHARED_MEMORY_SIZE 49152
#define MERGE_THREADBLOCK_SIZE 128

static uint *d_PartialHistograms;

/*
*	Function that maps value to bin in range 0 inclusive to binCOunt exclusive
*/
__global__ void clearHistogram(uint *d_Histogram, uint binCount)
{
//clear histogram
for (uint bin = UMAD(blockIdx.x, blockDim.x, threadIdx.x); bin < binCount; bin += UMUL(blockDim.x, gridDim.x))
d_Histogram[bin] = 0;
}