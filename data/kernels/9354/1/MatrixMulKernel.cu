#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// this amazingly nice error checking function is stolen from:
//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__global__ void MatrixMulKernel(double *OutMat, double *Mat1, double *Mat2,  int Arows, int Acols, int Bcols) {
// row and column within submatrix
int blockrow =  blockIdx.y;//*
int row = threadIdx.y;
int blockcol = blockIdx.x;
int col =  threadIdx.x ;

// allocate these arrays only once we can change the values in them later
__shared__ double subAshared[BLOCKSIZE*BLOCKSIZE];
__shared__ double subBshared[BLOCKSIZE*BLOCKSIZE];
double Cvalue=0;

for (int B = 0; B < ceil((double)(Acols / BLOCKSIZE)) + 1; B++) {
// fetch from global memory
// yes, these took a LONG time to figure out. Pencil and Paper FTW!

/* notice:
1) how these indexes are actually offset a multiple of B, *not 1*.
2) threads are offset by col which will be 1 apart for each thread
3) which means that means all threads in the warp are hitting successive global memory cells
*/
int Mat1index = (row + blockrow*BLOCKSIZE)*Acols + col + B*BLOCKSIZE;
int Mat2index = (B*BLOCKSIZE + row)*Bcols + BLOCKSIZE*blockcol + col;

if (Mat1index < Arows*Acols)
subAshared[row*BLOCKSIZE + col] = Mat1[Mat1index];
else
subAshared[row*BLOCKSIZE + col] = 0;

if (Mat2index < Acols*Bcols)
subBshared[row*BLOCKSIZE + col] = Mat2[Mat2index];
else
subBshared[row*BLOCKSIZE + col] = 0;

__syncthreads();

// this computation is all using shared memory (fast)
for (int j = 0; j < BLOCKSIZE; j++)
if ((row*BLOCKSIZE + j < BLOCKSIZE*BLOCKSIZE) && (j*BLOCKSIZE + col < BLOCKSIZE*BLOCKSIZE))
Cvalue += subAshared[row*BLOCKSIZE + j]*subBshared[j*BLOCKSIZE + col];

__syncthreads();

}
if ((row < Arows) && (col < Bcols)) {
int finalmatrow = blockrow*BLOCKSIZE + row;
int finalmatcol = blockcol*BLOCKSIZE + col;
OutMat[finalmatrow*Bcols +  finalmatcol] = Cvalue;
}
}