#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// this amazingly nice error checking function is stolen from:
//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__global__ void addKernel(double *c, const double *a, const double *b) {
int i = threadIdx.x;
c[i] = a[i] + b[i];
}