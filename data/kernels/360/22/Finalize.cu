#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Finalize(int size, int *originIn, int *originOut, int *bestSeenIn, int *bestSeenOut, int *adjIndexes, int *adjacency, int *mis, int *incomplete) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
int bestSeen = bestSeenIn[idx];
int origin = originIn[idx];
if (bestSeen < 1000001)
{
int start = adjIndexes[idx];
int end = adjIndexes[idx + 1];

// Look at all the neighbors and take best values:
for (int i = start; i < end; i++)
{
int neighbor = adjacency[i];
unsigned int challenger = bestSeenIn[neighbor];
int challengerOrigin = originIn[neighbor];

if (challenger > 0 && challenger == bestSeen && challengerOrigin > origin)
{
origin = challengerOrigin;
}

if (challenger > bestSeen)
{
bestSeen = challenger;
origin = challengerOrigin;
}
}
}

// Write new MIS status
int misStatus = -1;
if (origin == idx)
misStatus = 1;
else if (bestSeen == 1000001)
misStatus = 0;

mis[idx] = misStatus;


// If this node is still unassigned mark
if (misStatus == -1)
{
incomplete[0] = 1;
}
}
}