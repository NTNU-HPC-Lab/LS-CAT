#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getInducedGraphNeighborCountsKernel(int size, int *aggregateIdx, int *adjIndexesOut, int *permutedAdjIndexes, int *permutedAdjacencyIn)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int Begin = permutedAdjIndexes[ aggregateIdx[idx] ];
int End = permutedAdjIndexes[ aggregateIdx[idx + 1] ];

// Sort each section of the adjacency:
for(int i = Begin; i < End - 1; i++)
{
for(int ii = i + 1; ii < End; ii++)
{
if(permutedAdjacencyIn[i] < permutedAdjacencyIn[ii])
{
int temp = permutedAdjacencyIn[i];
permutedAdjacencyIn[i] = permutedAdjacencyIn[ii];
permutedAdjacencyIn[ii] = temp;
}
}
}

// Scan through the sorted adjacency to get the condensed adjacency:
int neighborCount = 1;
if(permutedAdjacencyIn[Begin] == idx)
neighborCount = 0;

for(int i = Begin + 1; i < End; i++)
{
if(permutedAdjacencyIn[i] != permutedAdjacencyIn[i - 1] && permutedAdjacencyIn[i] != idx)
{
permutedAdjacencyIn[neighborCount + Begin] = permutedAdjacencyIn[i];
neighborCount++;
}
}

// Store the size
adjIndexesOut[idx] = neighborCount;
}
}