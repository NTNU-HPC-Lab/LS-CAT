#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PreInitialize(int size, unsigned int *randoms, int *bestSeen, int *origin, int *mis) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Set the origin to be self
origin[idx] = idx;

// Set MIS to be -1;
mis[idx] = -1;

// Set the bestSeen value to be random
bestSeen[idx] = randoms[idx] % 1000000;
}
}