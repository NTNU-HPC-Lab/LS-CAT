#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MarkMerges(int size, int* desiredMerges, int* merging, int* mergesToMake, int* incomplete) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Find what aggregate this one wants to merge with
int desiredMerge = desiredMerges[idx];

// If this aggregate has a real potential merger:
if (desiredMerge >= 0)
{
// If the aggregates agree to merge mark as merging
if (desiredMerges[desiredMerge] == idx)
{
// Mark the merge as the higher indexed aggregate merging into lower
if (desiredMerge > idx)
mergesToMake[desiredMerge] =  idx;
else
mergesToMake[idx] = desiredMerge;

// Mark both aggregates as merging
merging[idx] = 1;
merging[desiredMerge] = 1;
}
// Otherwise mark incomplete to check again
else
{
incomplete[0] = 1;
}
}
}
}