#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillCondensedAdjacencyKernel(int size, int *aggregateIdx, int *adjIndexesOut, int *adjacencyOut, int *permutedAdjIndexesIn, int *permutedAdjacencyIn)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int oldBegin = permutedAdjIndexesIn[ aggregateIdx[idx] ];
int newBegin = adjIndexesOut[idx];
int runSize = adjIndexesOut[idx + 1] - newBegin;

// Copy adjacency over
for(int i = 0; i < runSize; i++)
{
adjacencyOut[newBegin + i] = permutedAdjacencyIn[oldBegin + i];
}
}
}