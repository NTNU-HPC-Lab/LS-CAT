#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MakeSplits_Large(int baseAggregateIdx, int* splitting, int* aggregation, int* aggMapAdjIndices, int* aggMapAdjacency, int* adjIndices, int* adjacency) {
int currentAgg = splitting[blockIdx.x];
int aggBegin = aggMapAdjIndices[currentAgg];
int aggSize = aggMapAdjIndices[currentAgg + 1] - aggBegin;
int newAgg = baseAggregateIdx + blockIdx.x;

__shared__ int nodeIds[256];
__shared__ int scratchA[256];
__shared__ int scratchB[256];
__shared__ int rootA;
__shared__ int rootB;
__shared__ int aCount, bCount;
__shared__ bool incomplete;
incomplete = true;
aCount = 1;
bCount = 1;

// Load in the node Id's from the aggregate map to the shared array:
if (threadIdx.x < aggSize)
nodeIds[threadIdx.x] = aggMapAdjacency[aggBegin + threadIdx.x];
__syncthreads();

// Each thread loads it's neighbors list into registers, translating into
// aggregate offsets.
int neighbors[40];
int nextNeighbor = 0;
int nodeId = -1;
if (threadIdx.x < aggSize)
{
nodeId = nodeIds[threadIdx.x];
int start = adjIndices[nodeId];
int end = adjIndices[nodeId + 1];
for (int i = start; i < end; i++)
{
int neighborId = adjacency[i];
int a = 0, b = aggSize - 1, midpoint;
while (a < b)
{
midpoint = a + ((b - a) / 2);
if (nodeIds[midpoint] < neighborId)
a = midpoint + 1;
else
b = midpoint;
}

if (nodeIds[a] == neighborId)
{
neighbors[nextNeighbor++] = a;
}
}
}
__syncthreads();

// Find the farthest node from the lowest indexed node (first root point)
// Start by marking the first node and threads without a node as seen
// Mark initial distances in scratch vector
if (threadIdx.x < aggSize)
scratchA[threadIdx.x] = threadIdx.x == 0 ? 0 : -1;
int myDist = threadIdx.x == 0 ? 0 : -1;
bool swapped = false;
incomplete = true;
__syncthreads();

while (incomplete)
{
// Set the incomplete flag to false
incomplete = false;
__syncthreads();

// Check if a neighbor has a positive distance
if (threadIdx.x < aggSize && myDist == -1)
{
for (int i = 0; i < nextNeighbor; i++)
{
int neighborDist = scratchA[neighbors[i]];
if (neighborDist > -1)
myDist = neighborDist + 1;
}
}
__syncthreads();

if (threadIdx.x < aggSize && myDist > 0 && !swapped)
{
swapped = true;
scratchA[threadIdx.x] = myDist;
rootA = threadIdx.x;
incomplete = true;
}
__syncthreads();
}

// Find the farthest node from the first root point (second root point)
// Start by marking the first node and threads without a node as seen
// Mark initial distances in scratch vector
if (threadIdx.x < aggSize)
scratchA[threadIdx.x] = threadIdx.x == rootA ? 0 : -1;
myDist = threadIdx.x == rootA ? 0 : -1;
swapped = false;
incomplete = true;
__syncthreads();

while (incomplete)
{
// Set the incomplete flag to false
incomplete = false;
__syncthreads();

// Check if a neighbor has a positive distance
if (threadIdx.x < aggSize && myDist == -1)
{
for (int i = 0; i < nextNeighbor; i++)
{
int neighborDist = scratchA[neighbors[i]];
if (neighborDist > -1)
{
myDist = neighborDist + 1;
}
}
}
__syncthreads();

if (threadIdx.x < aggSize && myDist > 0 && !swapped)
{
swapped = true;
scratchA[threadIdx.x] = myDist;
rootB = threadIdx.x;
incomplete = true;
}
__syncthreads();
}

// Setting an assigned aggregate label (In ScratchA) for every node with the node at
// rootA being assigned the current aggregate ID and the node at rootB
// being assigned the newAgg ID and set initial distances from a root node
// (In ScratchB) for each node, -1 for unknown and 0 for the roots
int myAggregate = -1;
if (threadIdx.x == rootA)
myAggregate = currentAgg;
if (threadIdx.x == rootB)
myAggregate = newAgg;
if (threadIdx.x < aggSize)
{
scratchA[threadIdx.x] = myAggregate;
scratchB[threadIdx.x] = myAggregate > -1 ? 0 : -1;
}
incomplete = true;
__syncthreads();

// Assign nodes to each aggregate until no unassigned nodes remain.
while (incomplete)
{
// Set the incomplete flag to false
incomplete = false;
__syncthreads();

if (threadIdx.x < aggSize && myAggregate == -1)
{
for (int i = 0; i < nextNeighbor; i++)
{
int neighborAgg = scratchA[neighbors[i]];
if (neighborAgg > -1)
{
myDist = scratchB[neighbors[i]] + 1;
myAggregate = neighborAgg;
}
}
if (myAggregate == -1)
incomplete = true;
if (myAggregate == newAgg)
atomicAdd((unsigned int*)&bCount, (unsigned)1);
if (myAggregate == currentAgg)
atomicAdd((unsigned int*)&aCount, (unsigned)1);
}
__syncthreads();

if (threadIdx.x < aggSize)
{
scratchA[threadIdx.x] = myAggregate;
scratchB[threadIdx.x] = myDist;
}
__syncthreads();
}

// If the split was uneven try to repair it
int sizeDifference = aCount > bCount ? aCount - bCount : bCount - aCount;
bool moveToA = aCount < bCount;
__shared__ int moved;
moved = 0;
int toMove = sizeDifference / 2;
incomplete = true;
__syncthreads();

while (incomplete && moved < toMove)
{
incomplete = false;
__syncthreads();
bool swapping = false;
int newDist = INT_MAX;
if (threadIdx.x < aggSize)
{
bool canSwap = moveToA ? myAggregate == newAgg : myAggregate == currentAgg;
bool borderNode = false;

// Check if this node has no same aggregate neighbors of higher distance
// and on a border
for (int i = 0; i < nextNeighbor; i++)
{
int neighborAgg = scratchA[neighbors[i]];
int neighborDist = scratchB[neighbors[i]];
if (neighborAgg == myAggregate && neighborDist > myDist)
canSwap = false;

if (neighborAgg != myAggregate)
{
if (neighborDist + 1 < newDist)
newDist = neighborDist + 1;
borderNode = true;
}
}

// If a node could swap see if it will
if (borderNode && canSwap && atomicAdd((unsigned int*)&moved, 1) < toMove)
{
swapping = true;
}
}
__syncthreads();
if (swapping)
{
int a = moveToA ? 1 : -1;
atomicAdd((unsigned int*)&bCount, -a);
atomicAdd((unsigned int*)&aCount, a);
scratchA[threadIdx.x] = moveToA ? currentAgg : newAgg;
scratchB[threadIdx.x] = newDist;
incomplete = true;
}
__syncthreads();
}

// Write out the values to the aggregation array
if (threadIdx.x < aggSize)
{
aggregation[nodeId] = scratchA[threadIdx.x];
}

}