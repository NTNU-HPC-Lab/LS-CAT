#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindDesirableMerges(int size, int minSize, int maxSize, bool force, int* adjIndices, int* adjacency, int *partSizes, int* desiredMerges, int* merging) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Only evaluate if the aggregate is not marked as active (merging
// or no possible merges)
if (merging[idx] != 1)
{
// Check through all neighboring aggregates for most desirable
int currentSize = partSizes[idx];
int checkedNeighbors = 0;
float bestDesirability = 0;
int mostDesirable = -1;
int start = adjIndices[idx];
int end = adjIndices[idx + 1];
for (int i = start; i < end; i++)
{
int neighborAgg = adjacency[i];

// Only active neighbor aggregates should be looked at:
if (merging[neighborAgg] != 1)
{
checkedNeighbors++;
int neighborSize = partSizes[neighborAgg];

float desirability = 0;
desirability += currentSize < minSize ? minSize - currentSize : 0;
desirability += neighborSize < minSize ? minSize - neighborSize : 0;
int totalSize = currentSize + neighborSize;
if (totalSize > maxSize)
desirability *= force ? 1.0/(totalSize - maxSize) : 0;

// If this merge is the most desirable seen mark it
if (desirability > bestDesirability)
{
bestDesirability = desirability;
mostDesirable = neighborAgg;
}
}
}

if (mostDesirable == -1)
merging[idx] = 1;

if (currentSize < minSize && force && mostDesirable == -1)
printf("Aggregate %d is too small but found no merges! %d / %d neighbors checked.\n",idx, checkedNeighbors, end-start);

desiredMerges[idx] = mostDesirable;
}
}
}