#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void removeRuntyPartsKernel(int size, int *partition, int *removeStencil, int *subtractions)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int currentNode = partition[idx];
if(removeStencil[currentNode] == 1)
partition[idx] = -1;
else
partition[idx] -= subtractions[currentNode];
}
}