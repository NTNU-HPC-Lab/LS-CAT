#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getInversePermutationKernel(int size, int *original, int *inverse)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
inverse[original[idx]] = idx;
}
}