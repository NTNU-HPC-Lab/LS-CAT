#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Initialize(int size, unsigned int *randoms, int *bestSeen, int *origin, int *mis, int *incomplete) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Taustep is performed with S1=13, S2=19, S3=12, and M=UINT_MAX coded into kernel
unsigned int z = randoms[idx];
unsigned int b = (((z << 13) ^ z) >> 19);
z = (((z & UINT_MAX) << 12) ^ b);

// Set the origin to be self
origin[idx] = idx;

// Set the bestSeen value to be either random from 0-1000000 or 1000001 if in MIS
int status = mis[idx];
int value = 0;
if (status == 1)
value = 1000001;

bestSeen[idx] = (mis[idx] == -1) ? (z % 1000000) : value;

// Write out new random value for seeding
randoms[idx] = z;
}

// Reset incomplete value
if (idx == 0)
incomplete[0] = 0;
}