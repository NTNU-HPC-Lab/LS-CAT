#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void packcoo_kernel(int num_entries, int* row_indices, int* column_indices, int* aggridx, int* partidx, int* partlabel)
{
int entryidx = blockIdx.x * blockDim.x + threadIdx.x;
if(entryidx < num_entries)
{
int row = row_indices[entryidx];
int col = column_indices[entryidx];
int l = partlabel[row];
int partstart = aggridx[partidx[l]];
unsigned int newindex = row - partstart;
newindex <<= 16;
newindex += col - partstart;
row_indices[entryidx] = newindex;
}
}