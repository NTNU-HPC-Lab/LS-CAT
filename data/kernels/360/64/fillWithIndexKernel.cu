#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillWithIndexKernel(int size, int *array)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
array[idx] = idx;
}
}