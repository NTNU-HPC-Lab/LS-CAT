#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void remapAggregateIdxKernel(int size, int *fineAggregateSort, int *aggregateRemapId)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
fineAggregateSort[idx] = aggregateRemapId[fineAggregateSort[idx]];
}
}