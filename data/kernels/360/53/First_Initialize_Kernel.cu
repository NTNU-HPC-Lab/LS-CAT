#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void First_Initialize_Kernel(int size, unsigned int *randoms, int *bestSeen, int *origin)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Set the origin to be self
origin[idx] = idx;

// Set the bestSeen value to be random
bestSeen[idx] = randoms[idx] % 1000000;
}
}