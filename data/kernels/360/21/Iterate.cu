#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Iterate(int size, int *originIn, int *originOut, int *bestSeenIn, int *bestSeenOut, int *adjIndexes, int *adjacency) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
int bestSeen = bestSeenIn[idx];
int origin = originIn[idx];
if (bestSeen < 1000001)
{
int start = adjIndexes[idx];
int end = adjIndexes[idx + 1];

// Look at all the neighbors and take best values:
for (int i = start; i < end; i++)
{
int neighbor = adjacency[i];
int challenger = bestSeenIn[neighbor];
int challengerOrigin = originIn[neighbor];

if (challenger > 0 && challenger == bestSeen && challengerOrigin > origin)
{
origin = challengerOrigin;
}


if (challenger > bestSeen)
{
bestSeen = challenger;
origin = challengerOrigin;
}
}
}

// Write out the best values found
bestSeenOut[idx] = bestSeen;
originOut[idx] = origin;
}
}