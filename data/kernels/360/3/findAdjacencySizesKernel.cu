#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findAdjacencySizesKernel(int size, int *adjIndexes, int *output) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
output[idx] = adjIndexes[idx + 1] - adjIndexes[idx];
}
}