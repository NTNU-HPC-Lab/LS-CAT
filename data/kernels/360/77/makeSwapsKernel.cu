#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeSwapsKernel(int size, int *partition, int *partSizes, int *nodeWeights, int *swap_to, int *swap_from, int *swap_index, float *desirability)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx == size - 1)
{
if(desirability[idx] > .1)
{
int swapTo = swap_to[idx];
int swapFrom = swap_from[idx];
int swapIndex = swap_index[idx];
int nodeWeight = nodeWeights[swapIndex];
partition[swapIndex] = swapTo;
atomicAdd(&partSizes[swapTo], nodeWeight);
atomicAdd(&partSizes[swapFrom], -nodeWeight);
//printf("Swapping node: %d, %d from part: %d, %d to part: %d, %d desirability: %f\n", swapIndex, nodeWeight, swapFrom, partSizes[swapFrom], swapTo, partSizes[swapTo], desirability[idx]);
}
}

else if(idx < size - 1)
{
if(desirability[idx] > .1 && swap_from[idx] != swap_from[idx + 1])
{
int swapTo = swap_to[idx];
int swapFrom = swap_from[idx];
int swapIndex = swap_index[idx];
int nodeWeight = nodeWeights[swapIndex];
partition[swapIndex] = swapTo;
atomicAdd(&partSizes[swapTo], nodeWeight);
atomicAdd(&partSizes[swapFrom], -nodeWeight);
//printf("Swapping node: %d, %d from part: %d, %d to part: %d, %d desirability: %f\n", swapIndex, nodeWeight, swapFrom, partSizes[swapFrom], swapTo, partSizes[swapTo], desirability[idx]);
}
}
}