#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindDesirableMergeSplits(int size, int minSize, int maxSize, int desiredSize, int* adjIndices, int* adjacency, int* partSizes, int* desiredMerges, int* merging) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Only evaluate if the aggregate is not marked as active (merging
// or no possible merges)
if (merging[idx] != 1)
{
// Check through all neighboring aggregates for most desirable
int currentSize = partSizes[idx];
int checkedNeighbors = 0;
bool currentOutSized = currentSize < minSize || currentSize > maxSize;
float bestDesirability = 0;
int mostDesirable = -1;
int start = adjIndices[idx];
int end = adjIndices[idx + 1];
for (int i = start; i < end; i++)
{
int neighborAgg = adjacency[i];

// Only active neighbor aggregates should be looked at:
if (merging[neighborAgg] != 1)
{
checkedNeighbors++;
int neighborSize = partSizes[neighborAgg];
bool neighborOutSized = neighborSize < minSize || neighborSize > maxSize;
int totalSize = currentSize + neighborSize;
bool legalPair = (neighborOutSized || currentOutSized) && totalSize > minSize * 2 && totalSize < maxSize * 2;
float desirability = legalPair ? 1.0 / abs(desiredSize - (currentSize + neighborSize)) : 0;

// If this merge is the most desirable seen mark it
if (desirability > bestDesirability)
{
bestDesirability = desirability;
mostDesirable = neighborAgg;
}
}
}

if (mostDesirable == -1)
merging[idx] = 1;

desiredMerges[idx] = mostDesirable;
}
}
}