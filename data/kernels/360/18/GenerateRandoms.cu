#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GenerateRandoms(int size, int iterations, unsigned int *randoms, unsigned int *seeds) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int z = seeds[idx];
int offset = idx;
int step = 32768;

for (int i = 0; i < iterations; i++)
{
if (offset < size)
{
unsigned int b = (((z << 13) ^ z) >> 19);
z = (((z & UINT_MAX) << 12) ^ b);
randoms[offset] = z;
offset += step;
}
}
}