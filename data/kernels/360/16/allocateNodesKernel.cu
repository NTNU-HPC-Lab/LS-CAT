#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void allocateNodesKernel(int size, int *adjIndexes, int *adjacency, int *partIn, int *partOut, int *aggregated) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
if (aggregated[idx] == 0)
{
int start = adjIndexes[idx];
int end = adjIndexes[idx + 1];

// Storage for possible aggregations.
int candidates[10];
int candidateCounts[10];
for (int i = 0; i < 10; i++)
{
candidates[i] = -1;
candidateCounts[i] = 0;
}

// Going through neighbors to aggregate:
for (int i = start; i < end; i++)
{
int candidate = partIn[adjacency[i]];
if (candidate != -1)
{
for (int j = 0; j < 10 && candidate != -1; j++)
{
if (candidates[j] == -1)
{
candidates[j] = candidate;
candidateCounts[j] = 1;
} else
{
if (candidates[j] == candidate)
{
candidateCounts[j] += 1;
candidate = -1;
}
}
}
}
}

// Finding the most adjacent aggregate and adding node to it:
int addTo = candidates[0];
int count = candidateCounts[0];
for (int i = 1; i < 10; i++)
{
if (candidateCounts[i] > count)
{
count = candidateCounts[i];
addTo = candidates[i];
}
}
partOut[idx] = addTo;
if (addTo != -1)
{
aggregated[idx] = 1;
}
}
}
}