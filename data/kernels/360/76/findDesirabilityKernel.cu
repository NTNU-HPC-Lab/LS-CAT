#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findDesirabilityKernel(int size, int optimalSize, int *adjIndexes, int *adjacency, int *partition, int *partSizes, int *nodeWeights, int *swap_to, int *swap_from, int *swap_index, float *desirability)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int currentPart = partition[idx];
int currentPartSize = partSizes[currentPart];
int nodeSize = nodeWeights[idx];
int selfAdjacency = 0;
int addTo = -1;
float bestDesirability = 0;

// The currentWeightFactor is higher the farther the count is from average
float currentWeightFactor = (float)abs(currentPartSize - optimalSize) / optimalSize;
// The self improvement is a measure of how much better this partitions size will be if the node is gone.
float selfImprovement = (abs(currentPartSize - optimalSize) - abs((currentPartSize - nodeSize) - optimalSize)) * currentWeightFactor;
if(selfImprovement > 0)
{
int start = adjIndexes[idx];
int end = adjIndexes[idx + 1];

// Arrays to store info about neighboring aggregates
int candidates[10];
int candidateCounts[10];
for(int i = 0; i < 10; i++)
{
candidates[i] = -1;
candidateCounts[i] = 0;
}

// Going through the neighbors:
for(int i = start; i < end; i++)
{
int candidate = partition[ adjacency[i] ];
if(candidate == currentPart)
selfAdjacency++;
else
for(int j = 0; j < 10; j++)
{
if(candidate != -1 && candidates[j] == -1)
{
candidates[j] = candidate;
candidateCounts[j] = 1;
candidate = -1;
}
else if(candidates[j] == candidate)
{
candidateCounts[j] += 1;
candidate = -1;
}
}
}

// Finding the best possible swap:
for(int i = 1; i < 10; i++)
{
if(candidates[i] != -1)
{
int neighborPart = candidates[i];
int neighborPartSize = partSizes[neighborPart];
float neighborWeightFactor = (float)abs(neighborPartSize - optimalSize) / optimalSize;
float neighborImprovement = ((float)(abs(neighborPartSize - optimalSize) - abs((neighborPartSize + nodeSize) - optimalSize))) * neighborWeightFactor;
// Combining with self improvement to get net
neighborImprovement += selfImprovement;
// Multiplying by adjacency factor
neighborImprovement *= (float)candidateCounts[i] / selfAdjacency;

if(neighborImprovement > bestDesirability)
{
addTo = neighborPart;
bestDesirability = neighborImprovement;
}
}
}
}

swap_from[idx] = currentPart;
swap_index[idx] = idx;
swap_to[idx] = addTo;
desirability[idx] = bestDesirability;
}
}