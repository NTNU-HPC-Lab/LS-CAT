#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void permuteInitialAdjacencyKernel(int size, int *adjIndexesIn, int *adjacencyIn, int *permutedAdjIndexesIn, int *permutedAdjacencyIn, int *ipermutation, int *fineAggregate)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int oldBegin = adjIndexesIn[ipermutation[idx]];
int oldEnd = adjIndexesIn[ipermutation[idx] + 1];
int runSize = oldEnd - oldBegin;
int newBegin = permutedAdjIndexesIn[idx];
//int newEnd = permutedAdjIndexesIn[idx + 1];
//int newRunSize = newEnd - newBegin;

//printf("Thread %d is copying from %d through %d into %d through %d\n", idx, oldBegin, oldEnd, newBegin, newEnd);

// Transfer old adjacency into new, while changing node id's with partition id's
for(int i = 0; i < runSize; i++)
{
permutedAdjacencyIn[newBegin + i] = fineAggregate[ adjacencyIn[oldBegin + i] ];
}
}
}