#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MarkSplits(int size, bool force, int minPartSize, int maxPartSize, int* partSizes, int* splitsToMake) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
int currentSize = partSizes[idx];
bool shouldSplit = currentSize > maxPartSize && (force || currentSize > minPartSize * 2);
splitsToMake[idx] = shouldSplit ? 1 : 0;
}
}