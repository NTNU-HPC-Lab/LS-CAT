#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulatedPartSizesKernel(int size, int *part, int *weights, int *accumulatedSize)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx == size - 1)
accumulatedSize[part[idx]] = weights[idx];
if(idx < size - 1)
{
int thisPart = part[idx];
if(thisPart != part[idx + 1])
accumulatedSize[thisPart] = weights[idx];
}
}