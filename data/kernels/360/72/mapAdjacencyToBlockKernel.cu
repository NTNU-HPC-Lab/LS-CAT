#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapAdjacencyToBlockKernel(int size, int *adjIndexes, int *adjacency, int *adjacencyBlockLabel, int *blockMappedAdjacency, int *fineAggregate)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
int begin = adjIndexes[idx];
int end = adjIndexes[idx + 1];
int thisBlock = fineAggregate[idx];

// Fill block labeled adjacency and block mapped adjacency vectors
for(int i = begin; i < end; i++)
{
int neighbor = fineAggregate[adjacency[i]];

if(thisBlock == neighbor)
{
adjacencyBlockLabel[i] = -1;
blockMappedAdjacency[i] = -1;
}
else
{
adjacencyBlockLabel[i] = thisBlock;
blockMappedAdjacency[i] = neighbor;
}
}
}
}