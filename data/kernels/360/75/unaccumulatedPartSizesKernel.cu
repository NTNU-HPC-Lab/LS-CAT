#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unaccumulatedPartSizesKernel(int size, int *accumulatedSize, int *sizes)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx == 0)
sizes[idx] = accumulatedSize[0];

else if(idx < size)
{
sizes[idx] = accumulatedSize[idx] - accumulatedSize[idx - 1];
}
}