#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findPartIndicesKernel(int size, int *array, int *partIndices) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
int value = array[idx];
int nextValue = (idx != size - 1) ? array[idx + 1] : -1;
if (value != nextValue)
{
partIndices[value + 1] = idx + 1;
}
}
}