#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillPartitionLabelKernel(int size, int *coarseAggregate, int *fineAggregateSort, int *partitionLabel)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
partitionLabel[idx] = coarseAggregate[ fineAggregateSort[idx] ];
}
}