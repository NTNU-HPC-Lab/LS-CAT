#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MakeMerges_MarkSplits(int size, int* mergeWith, int* offsets, int* mis, int* splitsToMake) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
int currentAgg = mis[idx];
int newAgg = mergeWith[currentAgg];
// If the aggregate is not merging just apply offset
if (newAgg == -1)
{
mis[idx] = currentAgg - offsets[currentAgg];
}
// The aggregate is merging find offset of aggregate merging with
else
{
int newId = newAgg - offsets[newAgg];
mis[idx] = newId;
splitsToMake[newId] = 1;
}
}
}