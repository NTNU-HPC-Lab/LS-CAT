#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getAggregateStartIndicesKernel(int size, int *fineAggregateSort, int *aggregateRemapIndex)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < size)
{
if(idx == 0 || fineAggregateSort[idx] != fineAggregateSort[idx - 1])
{
aggregateRemapIndex[fineAggregateSort[idx]] = idx;
}
}
}