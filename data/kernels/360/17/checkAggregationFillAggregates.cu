#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkAggregationFillAggregates(int size, int *adjIndices, int *adjacency, int* aggregation, int* valuesIn, int* valuesOut, int* incomplete) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < size)
{
// Find the currently marked distance
int currentVal = valuesIn[idx];
int currentAgg = aggregation[idx];

// Checking if any neighbors have a better value
int start = adjIndices[idx];
int end = adjIndices[idx + 1];
for (int i = start; i < end; i++)
{
int neighborAgg = aggregation[adjacency[i]];
int neighborVal = valuesIn[adjacency[i]];
if (neighborAgg == currentAgg && neighborVal > currentVal)
{
currentVal = neighborVal;
incomplete[0] = 1;
}
}

// Write out the distance to the output vector:
valuesOut[idx] = currentVal;
}
}