#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findPartIndicesNegStartKernel(int size, int *array, int *partIndices)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
if(idx < size)
{
int value = array[idx];
int nextValue = array[idx + 1];
if(value != nextValue)
partIndices[value + 1] = idx;
}
}