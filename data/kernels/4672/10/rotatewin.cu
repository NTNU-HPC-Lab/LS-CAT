#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotatewin(float* aframe2, float *aframe, float *win, int N, int offset){
int k = threadIdx.x + blockIdx.x*blockDim.x;
aframe2[(k+offset)%N] = win[k]*aframe[k];
}