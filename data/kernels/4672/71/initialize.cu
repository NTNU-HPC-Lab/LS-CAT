#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize(float* matrix, float sr, int numFrames, int length) {
int frame = blockIdx.y*blockDim.y + threadIdx.y;
int chan = (blockIdx.x*blockDim.x + threadIdx.x) << 1;
if ((frame < numFrames) && (chan < length)) {
matrix[frame*length+chan] = 0.0f;
matrix[frame*length+chan+1] = chan * sr / (length-2);
}
}