#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update(float* original, float* newTE, float* current, int nhalf) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
if (i < nhalf) {
current[i] /= nhalf;
newTE[i] = (original[i] < current[i]) ? current[i] : original[i];   // LIKELY, THERE IS A PERFORMANCE LOSS HERE
}
}