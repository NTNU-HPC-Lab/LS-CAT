#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fixPVandGain(float* input, float* output, float gain, int length) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
int j = i<<1;
if (j < length) {
if (isnan(output[j]))   // LIKELY, THERE IS A PERFORMANCE LOSS HERE
output[j] = 0.0f;  // set to zero any invalid amplitude
if (output[j+1] == -1.0f) {   // LIKELY, THERE IS A PERFORMANCE LOSS HERE
output[j] = 0.0f;   // set to zero the amp related to any undefined frequency
}
else
output[j] *= gain;   // scale all amplitudes by the gain factor
}
if (j == 0) output[0] = input[0];   // keep original DC amplitude
if (j == length-2) output[length-2] = input[length-2];   // keep original Nyquist amplitude
}