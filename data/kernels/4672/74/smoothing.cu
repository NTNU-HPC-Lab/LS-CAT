#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smoothing(float* input, float* output, double alpha, double beta, int length) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
int j = i<<1;
if (j < length) {
output[j] = (float) (input[j] * (1.0 + alpha) - output[j] * alpha);
output[j+1] = (float) (input[j+1] * (1.0 + beta) - output[j+1] * beta);
}
}