#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void morph(float* output, float* input1, float* input2, float ampCoeff, float freqCoeff, int length) {
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = i<<1;
if (j  < length) {
output[j] = input1[j]*(1.0-ampCoeff) + input2[j]*(ampCoeff);
output[j+1] = input1[j+1]*(1.0-freqCoeff) + input2[j+1]*(freqCoeff);
}
}