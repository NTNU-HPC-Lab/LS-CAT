#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naivekernel(float* output, float* frameA, float* frameB, int chans) {
int i = threadIdx.x + blockDim.x * blockIdx.x;
int j = i<<1;
if (i < chans) {
int test = frameA[j] >= frameB[j];
if (test) {
output[j] = frameA[j];
output[j+1] = frameA[j+1];
}
else {
output[j] = frameB[j];
output[j+1] = frameB[j+1];
}
}
}