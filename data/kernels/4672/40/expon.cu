#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expon(float* env, int nhalf) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
if (i < nhalf) {
env[i] = exp(env[i]/nhalf);   // exponentiate
}
}