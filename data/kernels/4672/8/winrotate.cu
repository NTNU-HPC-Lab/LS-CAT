#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void winrotate(float* inframe2, float* inframe, float *win, int N, int offset){
int k = (threadIdx.x + blockIdx.x*blockDim.x);
inframe2[k] = win[k]*inframe[(k+offset)%N];
}