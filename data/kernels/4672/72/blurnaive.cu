#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blurnaive(float* matrix, float* output, int firstFrame, int numFrames, int frameCount, int max, int length){
// int frame = firstFrame + (blockIdx.y*blockDim.y+ threadIdx.y);
int chan = (blockIdx.x*blockDim.x+ threadIdx.x)<<1;
float amp = 0.0f;
float freq = 0.0f;
int frame;
if (chan < length) {
for (frame = firstFrame; frame != frameCount; frame = (frame + 1) % max) {
amp += matrix[frame*length+chan];
freq += matrix[frame*length+chan+1];
}
output[chan] = (float) (amp / numFrames);
output[chan+1] = (float) (freq / numFrames);
}
}