#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
int diff;
if (i < nhalf) {
diff = fabs(nonSmoothed[i] - smoothed[i]/nhalf);
mask[i] = (diff > 0.23) ? 1 : 0;   // WHAT THRESHOLD TO USE?? different behaviour as opposed to CPU version!
}
}