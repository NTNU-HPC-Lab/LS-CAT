#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void takeLog(float* input, float* env, int nhalf) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
int j = i<<1;
if (i < nhalf) {
env[i] = log(input[j] > 0.0 ? input[j] : 1e-20);   // take the log of the amplitudes
}
}