#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
int i = threadIdx.x + blockDim.x*blockIdx.x;
int k = i + nCoefs;
if (k < nhalf+2-nCoefs) {
cepstrum[k] = 0.0;   // kill all the cepstrum coefficients above nCoefs
}
}