#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float4toUchar4(float4 *inputImage, uchar4 *outputImage, int width, int height)
{
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

float4 pixelf = inputImage[offset];
uchar4 pixel;
pixel.x = (unsigned char) pixelf.x; pixel.y = (unsigned char) pixelf.y;
pixel.z = (unsigned char) pixelf.z; pixel.w = (unsigned char) pixelf.w;

outputImage[offset] = pixel;
}