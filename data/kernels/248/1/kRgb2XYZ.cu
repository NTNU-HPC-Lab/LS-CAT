#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void kRgb2XYZ(uchar4* inputImg, float4* outputImg, int width, int height)
{
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

uchar4 nPixel=inputImg[offset];

float _b=(float)nPixel.x/255.0;
float _g=(float)nPixel.y/255.0;
float _r=(float)nPixel.z/255.0;

float x=_r*0.412453	+_g*0.357580	+_b*0.180423;
float y=_r*0.212671	+_g*0.715160	+_b*0.072169;
float z=_r*0.019334	+_g*0.119193	+_b*0.950227;

float4 fPixel;
fPixel.x=x;
fPixel.y=y;
fPixel.z=z;

outputImg[offset]=fPixel;
}