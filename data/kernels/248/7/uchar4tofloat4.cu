#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uchar4tofloat4(uchar4 *inputImage, float4 *outputImage, int width, int height)
{
int offsetX = blockIdx.x * blockDim.x + threadIdx.x;
int offsetY = blockIdx.y * blockDim.y + threadIdx.y;

if (offsetX < width && offsetY < height)
{
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

uchar4 pixel = inputImage[offset];
float4 pixelf;
pixelf.x = pixel.x; pixelf.y = pixel.y;
pixelf.z = pixel.z; pixelf.w = pixel.w;

outputImage[offset] = pixelf;
}
}