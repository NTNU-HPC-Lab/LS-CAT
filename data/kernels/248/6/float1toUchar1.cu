#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float1toUchar1(float1 *inputImage, uchar1 *outputImage, int width, int height)
{
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

float1 pixelf = inputImage[offset];
uchar1 pixel;
pixel.x = (unsigned char) pixelf.x;

outputImage[offset] = pixel;
}