#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float1toUchar4(float1 *inputImage, uchar4 *outputImage, int width, int height)
{
int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

float1 pixelf = inputImage[offset];
uchar4 pixel;
pixel.x = (unsigned char) pixelf.x; pixel.y = (unsigned char) pixelf.x;
pixel.z = (unsigned char) pixelf.x; pixel.w = (unsigned char) pixelf.x;

outputImage[offset] = pixel;
}