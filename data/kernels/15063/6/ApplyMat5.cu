#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyMat5(float* input, float* output, float* matrix){
int id = threadIdx.x + blockDim.x * blockIdx.x;

for (int i = 0; i < 296; ++i){
float total = 0.0f;
total += input[id * 300 + i] * matrix[0];
total += input[id * 300 + i + 1] * matrix[1];
total += input[id * 300 + i + 2] * matrix[2];
total += input[id * 300 + i + 3] * matrix[3];
total += input[id * 300 + i + 4] * matrix[4];

total += input[id * 300 + i + 300 * 1] * matrix[5];
total += input[id * 300 + i + 300 * 1 + 1] * matrix[6];
total += input[id * 300 + i + 300 * 1 + 2] * matrix[7];
total += input[id * 300 + i + 300 * 1 + 3] * matrix[8];
total += input[id * 300 + i + 300 * 1 + 4] * matrix[9];

total += input[id * 300 + i + 300 * 2] * matrix[10];
total += input[id * 300 + i + 300 * 2 + 1] * matrix[11];
total += input[id * 300 + i + 300 * 2 + 2] * matrix[12];
total += input[id * 300 + i + 300 * 2 + 3] * matrix[13];
total += input[id * 300 + i + 300 * 2 + 4] * matrix[14];

total += input[id * 300 + i + 300 * 3] * matrix[15];
total += input[id * 300 + i + 300 * 3 + 1] * matrix[16];
total += input[id * 300 + i + 300 * 3 + 2] * matrix[17];
total += input[id * 300 + i + 300 * 3 + 3] * matrix[18];
total += input[id * 300 + i + 300 * 3 + 4] * matrix[19];

total += input[id * 300 + i + 300 * 4] * matrix[20];
total += input[id * 300 + i + 300 * 4 + 1] * matrix[21];
total += input[id * 300 + i + 300 * 4 + 2] * matrix[22];
total += input[id * 300 + i + 300 * 4 + 3] * matrix[23];
total += input[id * 300 + i + 300 * 4 + 4] * matrix[24];

total = fmax(0.0f, total);

output[i + id * 296] = total;
}
}