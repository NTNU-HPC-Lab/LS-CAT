#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CombineScreen(float* d_postEdge1, float* d_postEdge2, float* d_postGradient1, float* d_postGradient2, float* d_postGradient3, float* d_postSobel3LR, float* d_postSobel3UD, float* d_postSmooth31, float* d_output){
int id = threadIdx.x + blockDim.x * blockIdx.x;

for (int i = 0; i < 73; ++i){
d_output[i + id * 73 + 73 * 73 * 0] =     d_postEdge1[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 1] =     d_postEdge2[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 2] = d_postGradient1[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 3] = d_postGradient2[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 4] = d_postGradient3[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 5] =  d_postSobel3LR[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 6] =  d_postSobel3UD[id * 73 + i];
d_output[i + id * 73 + 73 * 73 * 7] =  d_postSmooth31[id * 73 + i];
}
}