#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FirstHidden(float* input, float* weight, float* bias, int d_numVotes, float* d_votes){
int id = threadIdx.x + blockDim.x * blockIdx.x;

float total = 0.0f;

//printf("Num Votes: %i", d_numVotes);

for (int i = 0; i < d_numVotes; ++i){
//if (weight[id*d_numVotes + i] > 0) printf("Weight higher than 0: %f", weight[id*d_numVotes + i]);
//if (input[i] > 0) printf("Input: %f ", input[i]);
//printf("Weight: %f\n", weight[id * d_numVotes + i]);
//printf("Input: %f, Weight: %f\n", input[i], weight[id * d_numVotes + i]);
float sig = input[i] * weight[id * d_numVotes + i];
total += sig;// (1 / (1 + exp(-sig)));
}

//total /= d_numVotes;

printf("total: %f, Bias: %f\n", total, bias[id]);
total += bias[id];
//printf("Total: %f\n", total);
//printf("Bias: %f\n", bias[id]);
total = (1 / (1 + exp(-total)));
//total = ((int)(total)) % 3;
//if (total < 0.1) printf("Total %i: %f\n", id, total);

//printf("Total: %f\n", total);
d_votes[id] = total;
}