#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateHiddenWeights(float* d_weights, float error, float lr, int keyPress, float* d_outputweights, int screenSize, int numHiddenNeurons, float* d_bias, float* firstFire){
int id = threadIdx.x + blockDim.x * blockIdx.x;

float totalChange = 0.0f;
for (int i = 0; i < screenSize; ++i){
//Output weights stride is numNeurons, keypress is index into that section
float change = error * lr *d_outputweights[id * numHiddenNeurons + keyPress] * (firstFire[id] * 2 - 1);
totalChange += change;

d_weights[id * screenSize + i] = d_weights[id * screenSize + i] + change;

d_weights[id * screenSize + i] = min(1.0f, d_weights[id * screenSize + i]);
d_weights[id * screenSize + i] = max(0.0f, d_weights[id * screenSize + i]);
}
float biasChange = totalChange * -0.5f;
//printf("TotalChange: %f", biasChange);
d_bias[id] = d_bias[id] + biasChange;
}