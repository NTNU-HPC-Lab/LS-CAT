#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GreyScreen(float* d_pixelsR, float* d_pixelsG, float* d_pixelsB, float* d_reducePixels, int numPixels){
int id = threadIdx.x + blockIdx.x * blockDim.x;
//printf("Test ID: %u ", numPixels);
if (id < numPixels){
d_reducePixels[id] = (d_pixelsR[id] + d_pixelsG[id] + d_pixelsB[id]) / 3;
//printf("Reduce Pixels ");
//printf("%f ", d_reducePixels[id]);
}
}