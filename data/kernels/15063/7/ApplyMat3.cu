#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyMat3(float* input, float* output, float* matrix){
int id = threadIdx.x + blockDim.x * blockIdx.x;

//for (int i = 0; i < 148 * 148; ++i){
//	if(input[i] > 0.1f) printf("Input above 0, %i", i);
//}

for (int i = 0; i < 146; ++i){
float total = 0.0f;

//if (input[id * 148 + i] > 0.1f) printf("Input above 0, %i", id * 148 + i);

total += input[id * 148 + i] * matrix[0];
total += input[id * 148 + i + 1] * matrix[1];
total += input[id * 148 + i + 2] * matrix[2];

total += input[id * 148 + i + 148 * 1] * matrix[3];
total += input[id * 148 + i + 148 * 1 + 1] * matrix[4];
total += input[id * 148 + i + 148 * 1 + 2] * matrix[5];

total += input[id * 148 + i + 148 * 2] * matrix[6];
total += input[id * 148 + i + 148 * 2 + 1] * matrix[7];
total += input[id * 148 + i + 148 * 2 + 2] * matrix[8];

//if (total < -0.1f || total > 0.1f) printf("Total: %f", total);

total = fmax(0.0f, total);

output[i + id * 146] = total;
}
}