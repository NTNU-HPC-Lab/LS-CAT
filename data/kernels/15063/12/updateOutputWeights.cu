#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateOutputWeights(float* d_weights, float error, float lr, int keypress, int numHiddenNeurons, float* outputTotals, int numInput){
int id = threadIdx.x + blockDim.x * blockIdx.x;

int index = numHiddenNeurons * keypress + id;

float certainty = 0.0f;
for (int i = 0; i < numInput; ++i){
certainty += outputTotals[i];
}
certainty = outputTotals[keypress] / certainty;
//printf("Certainty: %f\n", certainty);

//int isPositive = 1;// d_weights[index] * 105 - 52.5;
//isPositive = min(isPositive, 1);
//isPositive = max(-1, isPositive);
//if (isPositive == 0){
//	isPositive = -1;
//}
//if(isPositive == 0)	printf("IsPositive: %i", isPositive);

//TODO test removing weight
float change = error * lr * d_weights[index] * certainty;

//printf("Error: %f, LR: %f, Weight: %f Change: %f\n", error, lr, d_weights[index], change);
d_weights[index] = d_weights[index] + change;

//Clamp
d_weights[index] = min(1.0f, d_weights[index]);
d_weights[index] = max(0.0f, d_weights[index]);
}