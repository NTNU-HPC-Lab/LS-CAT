#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OutputLayer(float* hiddenVotes, float* weight, int d_numHiddenNodes, float* d_votes){
int id = threadIdx.x + blockDim.x * blockIdx.x;

float total = 0.0f;

for (int i = 0; i < d_numHiddenNodes; ++i){
//printf("Hidden Votes: %i\n", hiddenVotes[i]);
//printf("Hidden Votes: %f, Weight: %f\n", hiddenVotes[i], weight[id * d_numHiddenNodes + i]);
total += hiddenVotes[i] * weight[id * d_numHiddenNodes + i];
//printf("Weight: %f", weight[id * d_numHiddenNodes + i]);
//printf("\n");
}

d_votes[id] = total;
//printf("Votes: %f\n", d_votes[id]);
}