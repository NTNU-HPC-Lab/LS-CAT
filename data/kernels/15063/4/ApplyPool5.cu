#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyPool5(float* input, float* output){
int id = threadIdx.x + blockIdx.x * blockDim.x;

for (int i = 0; i < 148; ++i){
//float total = input[i * 2 + id * 2 * 296] +
//	input[i * 2 + 1 + id * 2 * 296] + input[i * 2 + id * 2 * 296 + 296] + input[i * 2 + 1 + id * 2 * 296 + 296];
//total /= 4;

float total = 0;
total = max(	   input[i * 2 + id * 2 * 296],
input[i * 2 + id * 2 * 296 + 1]);
total = max(total, input[i * 2 + id * 2 * 296 + 296]);
total = max(total, input[i * 2 + id * 2 * 296 + 296 + 1]);

//float total = ((float)i) / 148.0f; // input[i * 2 + id * 2 * 296];

//if (total < -0.1f){
//	printf("ApplyFirstPool total: %f\n", total);
//}
output[i + id * 148] = total;
output[i + id * 148] = 1 / (1 + exp(-(output[i + id * 148] * 2 - 1)));
}
}