#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcInput(float* screen, float* weight, float* d_Votes, int stride){

//Current implementation, idk if it works. Probably doesn't, but it is worth a try, I think.
int id = threadIdx.x + blockDim.x * blockIdx.x;

d_Votes[id] = 0;

d_Votes[id] += screen[id] * weight[id];
d_Votes[id] += screen[id + 1] * weight[id + 1];
d_Votes[id] += screen[stride] * weight[stride];
d_Votes[id] += screen[stride + 1] * weight[stride + 1];

d_Votes[id] /= 4;
}