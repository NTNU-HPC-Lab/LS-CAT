#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplySecondPool(float* input, float* output){
int id = threadIdx.x + blockIdx.x * blockDim.x;

for (int i = 0; i < 73; ++i){
//float total = input[i * 2 + id * 2 * 296] +
//	input[i * 2 + 1 + id * 2 * 296] + input[i * 2 + id * 2 * 296 + 296] + input[i * 2 + 1 + id * 2 * 296 + 296];
//total /= 4;

float total = 0;
total = max(	   input[i * 2 + id * 2 * 146],
input[i * 2 + id * 2 * 146 + 1]);
total = max(total, input[i * 2 + id * 2 * 146 + 146]);
total = max(total, input[i * 2 + id * 2 * 146 + 146 + 1]);

output[i + id * 73] = total;//((float)i) / 73.0f;
output[i + id * 73] = 1 / (1 + exp(-(output[i + id * 73] * 2 - 1)));
}
}