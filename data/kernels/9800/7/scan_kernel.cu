#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_kernel(unsigned int* output_block, unsigned int block_num) {
__shared__ unsigned int shared_output[BLOCK_SIZE];

if (threadIdx.x >= block_num || threadIdx.x == 0) {
shared_output[threadIdx.x] = 0x0;
}  else {
shared_output[threadIdx.x] = output_block[threadIdx.x - 1];
}
__syncthreads();

for (unsigned int i = 1; i < block_num; i <<= 1) {
unsigned int val = 0;
if (threadIdx.x >= i) {
val = shared_output[threadIdx.x - i];
}
__syncthreads();
shared_output[threadIdx.x] += val;
__syncthreads();
}

if (threadIdx.x < block_num) {
output_block[threadIdx.x] = shared_output[threadIdx.x];
}
__syncthreads();
}