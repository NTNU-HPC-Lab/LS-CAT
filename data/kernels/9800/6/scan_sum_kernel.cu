#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_sum_kernel(unsigned int* input_vals, unsigned int pass, unsigned int * output, unsigned int* output_block, unsigned int size, unsigned int block_num) {
unsigned int tid = threadIdx.x;
unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
__shared__ unsigned int shared_input_vals[BLOCK_SIZE];
__shared__ unsigned int shared_output[BLOCK_SIZE];
if (mid >= size) {
shared_input_vals[tid] = 0xFFFFFFFF;
} else {
shared_input_vals[tid] = input_vals[mid];
}

__syncthreads();

if (tid == 0 || ((shared_input_vals[tid - 1] >> pass) & 0x01)) {
shared_output[tid] = 0;
} else {
shared_output[tid] = 1;
}

__syncthreads();

for (unsigned int i = 1; i < BLOCK_SIZE; i <<= 1) {
unsigned int val = 0;
if (tid >= i) {
val = shared_output[tid - i];
}
__syncthreads();
shared_output[tid] += val;
__syncthreads();
}

if (mid < size) {
output[mid] = shared_output[tid];
if ((mid == size - 1) || (tid == BLOCK_SIZE-1)) {
output_block[blockIdx.x] = shared_output[tid];
if (!((shared_input_vals[tid] >> pass) & 0x01)) {
//output_block[mid/BLOCK_SIZE] += 1;
output_block[blockIdx.x] += 1;
}
}
}
__syncthreads();
}