#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_large_sum_kernel(unsigned int* output_block, unsigned int* output_val, unsigned int* output_pos, unsigned int* input_val, unsigned int* input_pos, unsigned int* histogram, unsigned int pass, unsigned int block_num, unsigned int size) {

__shared__ unsigned int shared_prefix_sum[BLOCK_SIZE];
unsigned int tid = threadIdx.x;
unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;

if (mid >= size) {
shared_prefix_sum[tid] = 1;
} else {
shared_prefix_sum[tid] = output_block[blockIdx.x] + output_val[mid];
}
//if (shared_prefix_sum[tid] >= size) printf("mid/BLOCK_SIZE=%d\n", mid/BLOCK_SIZE);
__syncthreads();


if (mid < size) {
unsigned int location = shared_prefix_sum[tid];
if ((input_val[mid] >> pass) & 0x01) {
location = mid + histogram[0] - shared_prefix_sum[tid];
}
if (location >= size) printf("pass=%d,input[mid]=%d,mid=%d, blockIdx.x=%d, histogram[0]=%d, shared_prefix_sum[tid]=%d\n",
pass, input_val[mid], mid, blockIdx.x, histogram[0], shared_prefix_sum[tid]);
output_val[mid] = location;
}
__syncthreads();
}