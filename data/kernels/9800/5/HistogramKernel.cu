#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HistogramKernel(unsigned int * input, unsigned int size, unsigned int* histogram, unsigned int pass) {
int mid = threadIdx.x + blockIdx.x * blockDim.x;
if (mid < size) {
atomicAdd(&histogram[(input[mid]>>pass) & 0x01], 1);
}
}