#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter_kernel(unsigned int* d_inputVals, unsigned int* d_inputPos, unsigned int* d_outputVals, unsigned int* d_outputPos, unsigned int* cu_outputVals, size_t numElems) {
//unsigned int tid = threadIdx.x;
unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int val;
if (mid < numElems) {
val = cu_outputVals[mid];
}

if (mid < numElems) {
d_outputVals[val] = d_inputVals[mid];
d_outputPos[val] = d_inputPos[mid];
}
__syncthreads();
}