#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void categoricalCrossEntropyCost(float *predictions, float *target, int size, float *cost){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size)
{
float partial_cost = target[index] * logf(predictions[index]);
atomicAdd(cost, -partial_cost / size);
}
}