#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dCategoricalCrossEntropyCost(float *predictions, float *target, float *dY, int size){

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size)
{
dY[index] = (predictions[index] - target[index]);
}
}