#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void operator_transpose_h(const float *in, float *out, int height, int width) {
__shared__ float tile[TILE_SIZE][TILE_SIZE];

int batch_idx = blockIdx.z;
in += batch_idx * height * width;
out += batch_idx * height * width;

int bx = blockIdx.y;
int by = blockIdx.x;
int tx = threadIdx.y;
int ty = threadIdx.x;

int row = bx * TILE_SIZE + tx;
int col = by * TILE_SIZE + ty;

if (row < height && col < width) {
// coalesced read from global mem, TRANSPOSED write into shared mem:
tile[tx][ty] = in[row * width + col];

__syncthreads();

// read from shared mem, coalesced write to global mem
out[col * height + row] = tile[tx][ty];
}
}