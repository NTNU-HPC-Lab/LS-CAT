#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void operator_sum_h(const float *input1, float *output, const int *input1_shape, int input1_dims, const int *temp_shape, int dim, int dim_stride, int size) {
extern __shared__ int shared[];

int index = blockIdx.x * blockDim.x + threadIdx.x;

if (index < size) {
int *loc = (int *)shared + threadIdx.x * input1_dims;

index2loc(index, temp_shape, input1_dims - 1, loc);
for (int i = input1_dims - 1; i > dim; i--) {
loc[i] = loc[i - 1];
}
loc[dim] = 0;
int base = loc2index(loc, input1_shape, input1_dims);

int length = input1_shape[dim];
double total = 0;
for (int i = 0; i < length; i++) {
total += input1[base + i * dim_stride];
}

output[index] = total;
}
}