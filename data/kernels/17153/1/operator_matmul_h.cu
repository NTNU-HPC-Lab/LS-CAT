#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void operator_matmul_h(const float *input1, const float *input2, float *output, int height, int k, int width, int broadcast) {
__shared__ float shared_input1[TILE_SIZE][TILE_SIZE];
__shared__ float shared_input2[TILE_SIZE][TILE_SIZE];

int batch_idx = blockIdx.z;
if (broadcast != 1) input1 += batch_idx * height * k;
if (broadcast != 2) input2 += batch_idx * k * width;
output += batch_idx * height * width;

int bx = blockIdx.y;
int by = blockIdx.x;
int tx = threadIdx.y;
int ty = threadIdx.x;

int row = bx * TILE_SIZE + tx;
int col = by * TILE_SIZE + ty;
float v = 0;

for (int i = 0; i < (int)(ceil((float)k / TILE_SIZE)); i++) {
if (i * TILE_SIZE + ty < k && row < height)
shared_input1[tx][ty] = input1[row * k + i * TILE_SIZE + ty];
else
shared_input1[tx][ty] = 0;

if (i * TILE_SIZE + tx < k && col < width)
shared_input2[tx][ty] = input2[(i * TILE_SIZE + tx) * width + col];
else
shared_input2[tx][ty] = 0;
__syncthreads();

for (int j = 0; j < TILE_SIZE; j++)
v += shared_input1[tx][j] * shared_input2[j][ty];
__syncthreads();
}

if (row < height && col < width) output[row * width + col] = v;
}