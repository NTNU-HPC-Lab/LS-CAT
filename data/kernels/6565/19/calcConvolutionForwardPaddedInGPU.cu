#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcConvolutionForwardPaddedInGPU( float *in, float *padded_in, int batch_size, int in_size_x, int in_size_y, int in_size_z, int padding)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if( id < batch_size * in_size_x * in_size_y * in_size_z ){
int in_index = id;

int x = id % in_size_x;
id /= in_size_x;
int y = id % in_size_y;
id /= in_size_y;
int z = id % in_size_z;
id /= in_size_z;
int b = id;

int pad_index = b * (in_size_z * (in_size_x + 2*padding) * (in_size_y + 2*padding) ) +
z * ((in_size_x + 2*padding) * (in_size_y + 2*padding)) +
(y+padding) * (in_size_x + 2*padding) +
(x+padding) ;

padded_in[pad_index] = in[in_index];
}
/* original code
for ( int b = 0; b < in.size.b; ++b ){
for ( int z = 0; z < in.size.z; ++z ){
for ( int y = 0; y < in.size.y; ++y ){
for ( int x = 0; x < in.size.x; ++x ){
padded_in( b, padding+x, padding+y, z ) = in( b, x, y, z );
}
}
}
}
*/
}