#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDetectObjectsForwardGPU(float *in, float *out, int in_size_x, int in_size_y, int in_size_z, int max_bounding_boxes, int max_classes )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

for( int i = 0; i < max_bounding_boxes; i=i+(4+max_classes)){
int index = id * (in_size_x * in_size_y * in_size_z) + i;
out[index  ] = 1.0f / (1.0f + exp( -in[index  ] )); // x: sigmoid
out[index+1] = 1.0f / (1.0f + exp( -in[index+1] )); // y: sigmoid
out[index+2] = exp( in[index+2] ); // w: exp
out[index+3] = exp( in[index+3] ); // h: exp
for( int c = 0; c < max_classes; ++c){
int index2 = id * (in_size_x * in_size_y * in_size_z) + i+4+c;
out[index2] = 1.0f / (1.0f + exp( -in[index2] )); // id: sigmoid
}
}

/* original
for(int b = 0; b < in.size.b; ++b ){
for( int i = 0; i < _max_bounding_boxes; i=i+(4+_max_classes)){
out( b, i  , 0, 0 ) = 1.0f / (1.0f + exp( -in( b, i  , 0, 0 ) )); // x: sigmoid
out( b, i+1, 0, 0 ) = 1.0f / (1.0f + exp( -in( b, i+1, 0, 0 ) )); // y: sigmoid
out( b, i+2, 0, 0 ) = exp( in( b, i+2, 0, 0 ) ); // w: exp
out( b, i+3, 0, 0 ) = exp( in( b, i+3, 0, 0 ) ); // h: exp
for( int c = 0; c < _max_classes; ++c){
out( b, i+4+c, 0, 0 ) = 1.0f / (1.0f + exp( -in( b, i+4+c , 0, 0 ) )); // id: sigmoid
}
}
}
*/
}