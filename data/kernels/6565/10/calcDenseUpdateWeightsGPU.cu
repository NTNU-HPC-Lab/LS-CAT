#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseUpdateWeightsGPU( float *weights, float *biases, float *gradients, float *dW, float *dB, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z, float learning_rate, float momentum )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if ( id < out_size_x ) {
int w_size_x = in_size_x*in_size_y*in_size_z;
// int w_size_y = out_size_x;

for( int h = 0; h < w_size_x; ++h ){
// int index = id * (w_size_x * w_size_y) + h;
int index = h * out_size_x + id;
weights[index] = weights[index] - learning_rate * dW[index];
}

biases[id] = biases[id] - learning_rate * dB[id];

for( int b = 0; b < batch_size; ++b ){
int index = (b * out_size_x + id) * 2;
gradients[index+1] = gradients[index] + gradients[index+1] * momentum;
}
}

/* original
for (int i=0; i<weigts_data_num; ++i){
weights.data[i] = weights.data[i] - lr * 	dW.data[i];
}

for (int i=0; i<out.size.x; ++i){
biases.data[i] = biases.data[i] - lr * 	dB.data[i];
}

for ( int i = 0; i < out.size.x * in.size.b; ++i ){
GradientObject& grad = gradients[ i ];
grad.grad_prev = (grad.grad + grad.grad_prev * _momentum);
}
*/
}