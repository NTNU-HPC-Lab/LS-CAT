#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int Rand(unsigned int randx)
{
randx = randx*1103515245+12345;
return randx&2147483647;
}
__global__ void setRandom(float *gpu_array, int N, int maxval )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if( id < N ){
gpu_array[id] = 1.0f / maxval * Rand(id) / float( RAND_MAX );
}
}