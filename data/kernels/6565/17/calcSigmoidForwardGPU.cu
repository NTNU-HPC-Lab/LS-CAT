#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSigmoidForwardGPU(float *in, float *out, int elements)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if( id < elements ){
float v = in[id];
v = 1.0f / (1.0f + exp( -v )); // sigmoid
out[id] = v;
}

/* original
for ( int i = 0; i < in_total_size; ++i ){
out.data[i] = activator_function(in.data[i]);
}
*/
}