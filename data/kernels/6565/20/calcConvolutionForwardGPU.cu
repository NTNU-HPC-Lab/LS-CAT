#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcConvolutionForwardGPU( float *out, float *padded_in, float *filters, int padded_in_size_x, int padded_in_size_y, int padded_in_size_z, int batch_size, int out_size_x, int out_size_y, int out_size_z, int kernel_size, int stride, int filter_size)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
int id_out = id;

if (id_out < batch_size * out_size_x * out_size_y * out_size_z) {
int x = id % out_size_x;
id /= out_size_x;
int y = id % out_size_y;
id /= out_size_y;
int filter = id % out_size_z;
id /= out_size_z;
int b = id;

int mapped_x = x * stride;
int mapped_y = y * stride;

float sum = 0.0;
for ( int z = 0; z < padded_in_size_z; ++z ){ // padded_in_size_z = in_size_z
for ( int j = 0; j < kernel_size; ++j ){
for ( int i = 0; i < kernel_size; ++i ){

int padded_in_index = b * (padded_in_size_x * padded_in_size_y * padded_in_size_z) + z * (padded_in_size_x * padded_in_size_y) + (mapped_y + j) * (padded_in_size_x) + (mapped_x + i);
int filter_index = z * (kernel_size * kernel_size) + j * kernel_size + i;

sum += filters[filter * filter_size + filter_index] * padded_in[padded_in_index];
}
}
}
out[id_out] = sum;
}

/* original code
for ( int b = 0; b < in.size.b; ++b ){
int filters_size = filters.size();
for ( int filter = 0; filter < filters_size; ++filter ){
TensorObject<float> filter_data = filters[filter];
for ( int y = 0; y < out.size.y; ++y ){
for ( int x = 0; x < out.size.x; ++x ){
TensorCoordinate mapped = map_to_input( { 0, (uint16_t)x, (uint16_t)y, 0 }, 0 );
float sum = 0;
for ( int z = 0; z < in.size.z; ++z ){
for ( int j = 0; j < kernel_size; ++j ){
for ( int i = 0; i < kernel_size; ++i ){
sum += filter_data( 0, i, j, z ) * padded_in( b, mapped.x + i, mapped.y + j, z );
}
}
}
out( b, x, y, filter ) = sum;
}
}
}
}*/
}