#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcRouteBackwardGPU( float *dz_in, float *dz, int in_size_x, int in_size_y, int in_size_z, int z_offset, int elements )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if( id < elements ){
int id_out = id;
int x = id % in_size_x;
id /= in_size_x;
int y = id % in_size_y;
id /= in_size_y;
int z = id % in_size_z;
id /= in_size_z;
int b = id;

int id_in = b * (in_size_z * in_size_x * in_size_y) + (z + z_offset) * (in_size_x * in_size_y) + y * (in_size_x) + x;
dz[id_out] += dz_in[id_in];
}
/*
for ( int b = 0; b < layer_dz.size.b; ++b ){
for ( int z = 0; z < layer_dz.size.z; ++z ){
for ( int y = 0; y < layer_dz.size.y; ++y ){
for ( int x = 0; x < layer_dz.size.x; ++x ){
layer_dz( b, x, y, z ) += dz_in( b, x, y, z_offset+z );
}
}
}
}
*/
}