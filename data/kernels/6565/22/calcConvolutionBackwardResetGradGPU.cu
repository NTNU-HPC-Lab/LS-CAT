#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcConvolutionBackwardResetGradGPU( float *filter_grads, int in_size_z, int kernel_size, int filter_size, int elements )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if ( id < elements ) {
int i = id % kernel_size;
id /= kernel_size;
int j = id % kernel_size;
id /= kernel_size;
int z = id % in_size_z;
id /= in_size_z;
int filter = id;

int filter_grad_index = (filter * (in_size_z * kernel_size * kernel_size) + z * (kernel_size * kernel_size) + j * kernel_size + i) * 2;
filter_grads[ filter_grad_index ] = 0;
}

/* original code
int k_end = filter_grads.size();
int kernel_size_2 = kernel_size * kernel_size;
int i_end = kernel_size_2 * in.size.z;
for ( int k = 0; k < k_end; ++k ){
for ( int i = 0; i < i_end ; ++i ){
filter_grads[k].data[i].grad = 0;
}
}
*/
}