#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSoftmaxMaxForwardGPU(float *array, float *max, int *mutex, int batch_size, int in_size_x, unsigned n)
{
unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int stride = gridDim.x * blockDim.x; // = in_size_x
unsigned int offset = 0;

// __shared__ float cache[ 32 ][ BLOCK ]; // this should be constant. batch_size * in_size_x actually
extern __shared__ float cache[];
// printf("index=%d, stride=%d, n=%d, gridDim.x=%d, blockDim.x=%d\n", index, stride, n, gridDim.x, blockDim.x);

float temp = -1.0;
while(index + offset < n){
temp = fmaxf(temp, array[index + offset]);
offset += stride;
}

// cache[threadIdx.x] = temp;
cache[index] = temp;
__syncthreads();

unsigned int prev_i = blockDim.x;
unsigned int i = blockDim.x / 2;
while ( i!=0 ){
if(threadIdx.x < i){
// cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
cache[index] = fmaxf(cache[index], cache[index + i]);
}
if(prev_i%2 != 0){
cache[(blockIdx.x * blockDim.x)] = fmaxf(cache[(blockIdx.x * blockDim.x)], cache[(blockIdx.x * blockDim.x) + prev_i-1]);
}
__syncthreads();
i /= 2;
}

if( threadIdx.x == 0 ){
while( atomicCAS(mutex, 0, 1) != 0 ); // atomic compare and swap.
// *max = fmaxf(*max, cache[0]);
*(max+blockIdx.x) = fmaxf(*(max+blockIdx.x), cache[blockIdx.x * blockDim.x + 0]);
atomicExch(mutex, 0); // atomic exchange.
}

/* original
for ( int b = 0; b < in.size.b; ++b ){
float max_v = 0.0;
for ( int i = 0; i < in.size.x; ++i ){
float v = in( b, i, 0, 0 );
if(v>max_v){
max_v = v;
}
}
}
*/
}