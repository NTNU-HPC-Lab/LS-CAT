#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSoftmaxDivForwardGPU(float *out, float *sum, int batch_size, int in_size_x, unsigned int n)
{
// int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index<n && *(sum + blockIdx.x)>0.0){
// out[id] = out[id] / *sum;
out[index] = out[index] / *(sum + blockIdx.x);
}

/* original
for ( int i = 0; i < in.size.x; ++i ){
out( b, i, 0, 0 ) = out( b, i, 0, 0 ) / sum;
}
*/
}