#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaFillArray( float *gpu_array, float val, int N )
{
int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if( i < N ){
gpu_array[i] = val;
}
}