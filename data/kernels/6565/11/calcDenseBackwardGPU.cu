#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseBackwardGPU( float *dz_in, float *dz, float *in, float *weights, float *biases, float *gradients, float *dW, float *dB, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z, float momentum, float decay )
{
int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
int id_out = id;
int id_in  = id / out_size_x;

if( id_out < batch_size * in_size_x * in_size_y * in_size_z * out_size_x ){

int n = id % out_size_x;
id /= out_size_x;
int i = id % in_size_x;
id /= in_size_x;
int j = id % in_size_y;
id /= in_size_y;
int z = id % in_size_z;
id /= in_size_z;
int b = id;

int w_size_x = in_size_x * in_size_y * in_size_z;
// int w_size_y = out_size_x;

int m = z * (in_size_x * in_size_y) + j * (in_size_x) + i;

//    for ( int n = 0; n < out_size_x; ++n ){
float dzin = dz_in[b * (out_size_x * out_size_y * out_size_z) + n];

int w_index = n * w_size_x + m;

float w = weights[w_index];
gradients[ (n*batch_size + b) * 2 ] = dzin;
dz[id_in] += dzin * w;

dW[w_index] += in[id_in] * (gradients[ (n*batch_size + b) * 2 ] + gradients[ (n*batch_size + b) * 2 + 1 ] * momentum) + (decay * w);
//    }

}

/* original
for ( int n = 0; n < out.size.x; ++n ){
for ( int z = 0; z < in.size.z; ++z ){
for ( int j = 0; j < in.size.y; ++j ){
for ( int i = 0; i < in.size.x; ++i ){
int m = map( { 0, i, j, z } );

for( int b = 0; b < in.size.b; ++b ){
GradientObject& grad = gradients[ n*in.size.b + b ];
float dzin = dz_in( b, n, 0, 0 );
float w = weights(0, m, n, 0);
grad.grad = dzin;

dz( b, i, j, z ) += dzin * w;
dW( 0, m, n, 0 ) += in( b, i, j, z ) * (grad.grad + grad.grad_prev * _momentum) + (_decay * w);
}
}
}
}
}
// original to here
*/
}