#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcConvolutionUpdateWeightsGPU( float *filters, float *filter_grads, int in_size_z, int number_filters, int kernel_size, float momentum, float decay, float learning_rate, int elements )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if ( id < elements ) {

int id_out = id;
int i = id % kernel_size;
id /= kernel_size;
int j = id % kernel_size;
id /= kernel_size;
int z = id % in_size_z;
id /= in_size_z;
int filter = id;

int filter_size = 1 * kernel_size * kernel_size * in_size_z;
int filter_grad_index = (filter * filter_size + z * (kernel_size * kernel_size) + j * kernel_size + i) * 2;

float grad = filter_grads[ filter_grad_index ];
float grad_prev = filter_grads[ filter_grad_index + 1 ];
float m = ( grad + grad_prev * momentum );

filter_grads[ filter_grad_index + 1 ] = m;

float w = filters[ id_out ];
w -= learning_rate * ( m + (decay * w));
filters[ id_out ] = w;
}

/* original code
int filters_size = filters.size();
for ( int a = 0; a < filters_size; ++a ){
for ( int z = 0; z < in.size.z; ++z ){
for ( int j = 0; j < kernel_size; ++j ){
for ( int i = 0; i < kernel_size; ++i ){
GradientObject& grad = filter_grads[a].get( 0, i, j, z );
float m = (grad.grad + grad.grad_prev * momentum);
grad.grad_prev = m;
float& w = filters[a].get( 0, i, j, z );
w -= lr * ( m + (decay * w));
}
}
}
}
*/
}