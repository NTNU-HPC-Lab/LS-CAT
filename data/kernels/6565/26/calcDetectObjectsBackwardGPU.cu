#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activator_derivative( float x )
{
float sig = 1.0f / (1.0f + exp( -x ));
return sig * (1 - sig);
}
__global__ void calcDetectObjectsBackwardGPU( float *dz_in, float *dz, float *in, int batch_size, int in_size_x, int in_size_y, int in_size_z, int max_bounding_boxes, int max_classes )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

for( int i = 0; i < max_bounding_boxes; i=i+(4+max_classes)){
int index = id * (in_size_x * in_size_y * in_size_z) + i;

dz[index  ] = activator_derivative( in[index  ] ) * dz_in[index  ]; // x: sigmoid derivative * grads
dz[index+1] = activator_derivative( in[index+1] ) * dz_in[index+1]; // y: sigmoid derivative * grads
dz[index+2] = exp( in[index+2] ) * dz_in[index+2]; // w: exp * grads
dz[index+3] = exp( in[index+3] ) * dz_in[index+3]; // w: exp * grads
for( int c = 0; c <max_classes; ++c){
int index2 = id * (in_size_x * in_size_y * in_size_z) + i+4+c;
dz[index2] = activator_derivative( in[index2] ) * dz_in[index2]; // id: sigmoid derivative * grads
}
}

/* original code
for(int b = 0; b < dz_in.size.b; ++b ){
for( int i = 0; i < _max_bounding_boxes; i=i+(4+_max_classes)){
dz( b, i  , 0, 0 ) = activator_derivative( in( b, i  , 0, 0 ) ) * dz_in( b, i  , 0, 0 ); // x: sigmoid derivative * grads
dz( b, i+1, 0, 0 ) = activator_derivative( in( b, i+1 , 0, 0 ) ) * dz_in( b, i+1, 0, 0 ); // y: sigmoid derivative * grads
dz( b, i+2, 0, 0 ) = exp( in( b, i+2, 0, 0 ) ) * dz_in( b, i+2, 0, 0 ); // w: exp * grads
dz( b, i+3, 0, 0 ) = exp( in( b, i+3, 0, 0 ) ) * dz_in( b, i+3, 0, 0 ); // h: exp * grads
for( int c = 0; c <_max_classes; ++c){
dz( b, i+4+c, 0, 0 ) = activator_derivative( in( b, i+4+c , 0, 0 ) ) * dz_in( b, i+4+c , 0, 0 ); // id: sigmoid derivative * grads
}
}
}
*/
}