#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcMaxPoolForwardGPU( float *in,float *out, int in_size_x, int in_size_y, int in_size_z, int batch_size, int out_size_x, int out_size_y, int out_size_z, int stride, int kernel_size )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
int id_out = id;

if( id_out < batch_size * out_size_x * out_size_y * out_size_z) {
int x = id % out_size_x;
id /= out_size_x;
int y = id % out_size_y;
id /= out_size_y;
int z = id % out_size_z;
id /= out_size_z;
int b = id;

int mapped_x = x * stride;
int mapped_y = y * stride;

float mval = -1000000.0;
for ( int j = 0; j < kernel_size; ++j ){
for ( int i = 0; i < kernel_size; ++i ){

int id_in = b * (in_size_z * in_size_x * in_size_y) +
z * (in_size_x * in_size_y) +
(mapped_y + j) * (in_size_x) +
(mapped_x + i);

float v = in[id_in];
if ( v > mval ){
mval = v;
}
}
}
out[id_out] = mval;
}

/* original
for ( int b = 0; b < in.size.b; ++b ){
for ( int z = 0; z < out.size.z; ++z ){
for ( int y = 0; y < out.size.y; ++y ){
for ( int x = 0; x < out.size.x; ++x ){
TensorCoordinate mapped = map_to_input( { 0, (uint16_t)x, (uint16_t)y, 0 }, 0 );
float mval = -FLT_MAX;
for ( int j = 0; j < kernel_size; ++j ){
for ( int i = 0; i < kernel_size; ++i ){
float v = in( b, mapped.x + i, mapped.y + j, z );
if ( v > mval ){
mval = v;
}
}
}
out( b, x, y, z ) = mval;
}
}
}
}

*/
}