#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseBarckwardNabraBGPU( float *dz_in, float *dB, int batch_size, int out_size_x ){
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if( id < out_size_x ){
for( int b = 0; b < batch_size; ++b ){
dB[id] += dz_in[ b * (out_size_x) + id ];
}
}
/* original
for ( int n = 0; n < out.size.x; ++n ){
for( int b = 0; b < in.size.b; ++b ){
dB( 0, 0, n, 0 ) += dz_in( b, n, 0, 0 );
}
}
*/
}