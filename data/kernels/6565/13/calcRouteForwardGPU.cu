#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcRouteForwardGPU(float *in, float *out, int in_size_x, int in_size_y, int in_size_z, int z_offset, int elements )
{
// int i = blockIdx.x*blockDim.x + threadIdx.x;
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if( id < elements ){
int id_in = id;

int x = id % in_size_x;
id /= in_size_x;
int y = id % in_size_y;
id /= in_size_y;
int z = id % in_size_z;
id /= in_size_z;
int b = id;

int id_out = b * (in_size_z * in_size_x * in_size_y) + (z + z_offset) * (in_size_x * in_size_y) + y * (in_size_x) + x;
out[id_out] = in[id_in];
}

/* original code
for ( int b = 0; b < layer_in.size.b; ++b ){
for ( int z = 0; z < layer_in.size.z; ++z ){
for ( int y = 0; y < layer_in.size.y; y++ ){
for ( int x = 0; x < layer_in.size.x; x++ ){
out( b, x, y, z_offset+z ) = layer_in( b, x, y, z );
}
}
}
}
*/

}