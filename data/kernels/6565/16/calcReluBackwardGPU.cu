#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcReluBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *in, int elements )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if( id < elements ){
dz_in[id] += dz_next_layer[id];
dz[id] += (in[id] < 0) ? (0) : (1.0 * dz_in[id]);
}

/* original
for( unsigned i = 0; i < data_size; ++i ){
dz_in.data[i] += dz_next_layer.data[i];
dz.data[i] +=  (in.data[i] < 0) ? (0) : (1.0 * dz_in.data[i]);
}
*/
}