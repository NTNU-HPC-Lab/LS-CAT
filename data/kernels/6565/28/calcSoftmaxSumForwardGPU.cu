#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSoftmaxSumForwardGPU(float *array, float *out, float *max, float *sum, int *mutex, int batch_size, int in_size_x, unsigned n)
{
unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int stride = gridDim.x * blockDim.x;
unsigned int offset = 0;

// __shared__ float cache[ 32 ][ BLOCK ]; // max 0xc000
extern __shared__ float cache[];

float temp = 0.0;
while(index + offset < n){
// float v = exp(array[index + offset] - *max);
float v = exp(array[index + offset] - *(max + blockIdx.x));
out[index + offset] = v;
temp = temp + v;
offset += stride;
}

// cache[threadIdx.x] = temp;
cache[index] = temp;

__syncthreads();

unsigned int prev_i = blockDim.x;
unsigned int i = blockDim.x / 2;

while ( i!=0 ){
if(threadIdx.x < i){
cache[index] = cache[index] + cache[index + i];
}
if(prev_i%2 != 0){
cache[blockIdx.x * blockDim.x + 0] = cache[blockIdx.x * blockDim.x + 0] + cache[blockIdx.x * blockDim.x + prev_i-1];
}
__syncthreads();
prev_i = i;
i /= 2;
}

if( threadIdx.x == 0 ){
while( atomicCAS(mutex, 0, 1) != 0 );
// *sum = *sum + cache[blockIdx.x][0];
*(sum+blockIdx.x) = *(sum+blockIdx.x) + cache[blockIdx.x * blockDim.x + 0];
atomicExch(mutex, 0);
}

/* original
float sum = 0.0;
for ( int i = 0; i < in.size.x; ++i ){
float v = in( b, i, 0, 0 );
v = exp(v - max_v);
out( b, i, 0, 0 ) = v;
sum += v;
}
*/
}