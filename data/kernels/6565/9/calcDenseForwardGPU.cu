#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseForwardGPU( float *in, float *out, float *weights, float *biases, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z )
{
int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
int id_out = id;
if ( id_out < batch_size * out_size_x * out_size_y * out_size_z ){
int n = id % out_size_x;
id /= out_size_x;
// int y = id % out_size_y;
id /= out_size_y;
// int z = id % out_size_z;
id /= out_size_z;
int b = id;

int w_size_x = in_size_x*in_size_y*in_size_z;

float sum = 0;
for ( int k = 0; k < in_size_z; ++k ){
for ( int j = 0; j < in_size_y; ++j ){
for ( int i = 0; i < in_size_x; ++i ){
int m = k * (in_size_x * in_size_y) + j * (in_size_x) + i;
int w_index = n * (w_size_x) + m;
int in_index = b * (in_size_x * in_size_y * in_size_z) + k * (in_size_x * in_size_y) + j * in_size_x + i;
sum += in[in_index] * weights[w_index];
}
}
}
int bias_index = n;
out[id_out] = sum + biases[bias_index];
}

/* original
for ( int b = 0; b < in.size.b; ++b ){
for ( int n = 0; n < out.size.x; ++n ){
float sum = 0;
for ( int z = 0; z < in.size.z; ++z ){
for ( int j = 0; j < in.size.y; ++j ){
for ( int i = 0; i < in.size.x; ++i ){
int m = map( { 0, i, j, z } );
sum += in( b, i, j, z ) * weights( 0, m, n, 0 );
}
}
}
out( b, n, 0, 0 ) = sum + biases( 0, 0, n, 0);
}
}
*/
}