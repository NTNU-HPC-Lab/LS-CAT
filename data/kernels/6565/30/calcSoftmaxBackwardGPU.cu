#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcSoftmaxBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, unsigned int n )
{
int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
// unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

if ( index < n ){
dz_in[index] += dz_next_layer[index];
dz[index] +=  dz_in[index];
}

/* original
for( int i = 0; i < dz_in.size.b * dz_in.size.x * dz_in.size.y * dz_in.size.z; ++i ){
dz_in.data[i] += dz_next_layer.data[i];
}

for ( int i = 0; i < in.size.b * in.size.x * in.size.y * in.size.z; ++i ){
dz.data[i] += dz_in.data[i];
}
*/
}