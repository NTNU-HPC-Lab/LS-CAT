#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activator_derivative( float x )
{
float sig = 1.0f / (1.0f + exp( -x ));
return sig * (1 - sig);
}
__global__ void calcSigmoidBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *in, int elements )
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

if( id < elements ){
float x = dz_in[id] += dz_next_layer[id];
float sig = 1.0f / (1.0f + exp( -x ));
dz[id] +=  ( sig * (1 - sig) ) * dz_in[id]; // sigmoid_derivative * dz_in
}

/* original
for( int i = 0; i < dz_in.size.b * dz_in.size.x * dz_in.size.y * dz_in.size.z; ++i ){
dz_in.data[i] += dz_next_layer.data[i];
}

for ( int i = 0; i < in_total_size; ++i ){
dz.data[i] += activator_derivative( in.data[i] ) * dz_in.data[i];
}
*/
}