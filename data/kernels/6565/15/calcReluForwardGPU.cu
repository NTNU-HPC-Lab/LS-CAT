#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcReluForwardGPU(float *in, float *out, int elements)
{
int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
if( id < elements ){
float v = in[id];
if ( v < 0 ){
v = 0.0;
}
out[id] = v;
}

/* original
for( unsigned i = 0; i < data_size; ++i ){
float v = in.data[i];
if ( v < 0 ){
v = 0;
}
out.data[i] = v;
}
*/
}