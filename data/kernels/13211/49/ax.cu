#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vsquare.cu
// a simple CUDA kernel to element multiply vector with itself

extern "C"   // ensure function name to be exactly "vsquare"
{
}
__global__ void ax(const int lengthC, const double a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthC)
{
c[i] = a*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}