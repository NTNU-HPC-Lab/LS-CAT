#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmultbang"
{
}
__global__ void gaxpy4(const int n, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
if (i < n) {
c[i] = (double) i;  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}

}