#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void tx1mx(const int lengthX, const double *t, const double *x,  double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] += t[i]*x[i]*(1.0-x[i]);
}
}