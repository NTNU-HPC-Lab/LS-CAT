#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void rectlin(const int n, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
if (a[i]>0.0)
{b[i] = a[i];}
else
{b[i] = 0.0;}
}
}