#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void xsigny_update(const int n, const double *a, double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
if (b[i]>0)
{c[i]+=a[i];}
else
{if (b[i]<0)
{c[i]-=a[i];}
}
}
}