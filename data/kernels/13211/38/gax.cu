#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void gax(const int lengthC, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthC)
{
c[i] = a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}