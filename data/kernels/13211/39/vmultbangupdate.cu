#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gax.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gax"
{
}
__global__ void vmultbangupdate(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
c[i] += alpha*a[i] * b[i];
}
}