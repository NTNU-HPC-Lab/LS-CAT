#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmult!"
{
}
__global__ void vdivbang(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
c[i] = alpha*a[i] / b[i];
}
}