#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"   // ensure function name to be exactly "gax"
{
}
__global__ void sigmoid(const int lengthA, const double *a,  double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
b[i] = exp(-a[i]);
b[i] = 1.0/(1.0+b[i]);
}
}