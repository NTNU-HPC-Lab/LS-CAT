#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vsquare.cu
// a simple CUDA kernel to element multiply vector with itself

extern "C"   // ensure function name to be exactly "vsquare"
{
}
__global__ void expkernel(const int lengthA, const double *a,  double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
b[i] = exp(a[i]);
}
}