#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vmult(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
c[i] = alpha*a[i] * b[i];
}
}