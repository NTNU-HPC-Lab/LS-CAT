#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gaxpy.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy"
{
}
__global__ void Dstanh(const int lengthX, const double sf, const double *gradc, const double *fc,  double *gradn)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
gradn[i] += sf*gradc[i]*(1.0-(fc[i]/sf)*(fc[i]/sf));
}
}