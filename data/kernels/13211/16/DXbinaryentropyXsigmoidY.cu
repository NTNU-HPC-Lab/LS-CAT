#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void DXbinaryentropyXsigmoidY(const int lengthX, const double *x,  const double *y, const double *t, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] -= t[0]*(y[i]-log(x[i]/(1.0-x[i])))/lengthX;
}
}