#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void DmeanSquareLoss(const int lengthx, const double pref, const double *gradc, const double *x,const double *y, double *gradn )
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthx)
{
gradn[i] += pref * gradc[0] * (x[i]-y[i]);
}
}