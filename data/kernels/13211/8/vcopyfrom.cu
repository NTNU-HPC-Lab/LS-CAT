#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vcopyfrom(const int n, const int shift, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
b[i] = a[i+shift];
}
}