#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmult!"
{
}
__global__ void logkernel(const int lengthA, const double *a,  double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
b[i] = log(a[i]);
}
}