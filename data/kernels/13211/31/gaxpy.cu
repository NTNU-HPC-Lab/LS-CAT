#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void gaxpy(const int lengthC, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthC)
{
c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}