#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void DYbinaryentropyXsigmoidY(const int lengthX, const double *x,  const double *y, const double *t, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] += t[0]*(1.0/(1.0+exp(-y[i]))-x[i])/lengthX;
}
}