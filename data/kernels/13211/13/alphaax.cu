#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void alphaax(const int lengthC, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthC)
{
c[i] = alpha*a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}