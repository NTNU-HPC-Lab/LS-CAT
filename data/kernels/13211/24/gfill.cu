#include "hip/hip_runtime.h"
#include "includes.h"
// B=diag(A)

extern "C"
{
}
__global__ void gfill(const int n, const double *a, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
c[i] = a[0];
}
}