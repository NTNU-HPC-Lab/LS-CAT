#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void DXbinaryentropy(const int lengthX, const double *x,  const double *y, const double *t, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] += t[0]*log(x[i]*(1.0-y[i])/(y[i]*(1.0-x[i])))/lengthX;
}
}