#include "hip/hip_runtime.h"
#include "includes.h"
// filename: ax.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "ax"
{
}
__global__ void CalpahGaxpGy(const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = alpha*a[0]*b[i]+c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}