#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void gscale(const int lengthB, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthB)
{
b[i] = a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}