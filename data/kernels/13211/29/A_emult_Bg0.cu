#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void A_emult_Bg0(const int n, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
if (b[i]>0.0)
{c[i] += a[i];}
else
{c[i] += 0.0;}
}
}