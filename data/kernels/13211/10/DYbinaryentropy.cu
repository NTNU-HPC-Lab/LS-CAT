#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void DYbinaryentropy(const int lengthX, const double *x,  const double *y, const double *t, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] += t[0]*((y[i]-x[i])/(y[i]*(1.0-y[i])))/lengthX;
}
}