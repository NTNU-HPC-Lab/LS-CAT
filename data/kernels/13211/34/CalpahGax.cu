#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
}
__global__ void CalpahGax(const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = alpha*a[0]*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}