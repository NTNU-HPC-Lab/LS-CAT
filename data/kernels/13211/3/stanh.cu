#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void stanh(const int lengthA, const double alpha, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
b[i] = alpha*tanh(a[i]);
}
}