#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void alphaaxpy(const int lengthC, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthC)
{
c[i] = alpha*a[0]*b[i]+c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}
}