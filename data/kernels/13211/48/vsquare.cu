#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gax.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gax"
{
}
__global__ void vsquare(const double *a, double *c)
{
int i = threadIdx.x+blockIdx.x*blockDim.x;
double v = a[i];
c[i] = v*v;
}