#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void gaxpy2(const double *a, const double *b, double *c)
{
int i = threadIdx.x + threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}