#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmultbang"
{
}
__global__ void binaryentropy(const int lengthX, const double *x,  const double *y, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i] = x[i]*log(x[i]/y[i])+ (1.0-x[i])*log((1.0-x[i])/(1.0-y[i]));
}
}