#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vadd(const int n, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
c[i] = a[i] + b[i];
}
}