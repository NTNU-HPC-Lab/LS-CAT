#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void gaxpy3(const int n, const double *a, const double *b, double *c)
{
int i = threadIdx.x + threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
if (i < n) {
printf("%d\n",i);
c[i] = a[0]*b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}

}