#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmult!"
{
}
__global__ void vabs(const int n, const double *a, double *b)
{
int i =	 threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{b[i]=fabs(a[i]);}
}