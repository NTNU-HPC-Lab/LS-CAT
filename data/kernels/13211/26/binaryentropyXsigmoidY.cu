#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
}
__global__ void binaryentropyXsigmoidY(const int lengthX, const double *x,  const double *y, double *z)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthX)
{
z[i]=x[i]*log(x[i])+(1.0-x[i])*log(1.0-x[i])-x[i]*y[i]+log(1.0+exp(y[i]));
}
}