#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vAoverBupdate(const int lengthA, const double alpha, const double *gradc, const double *a, const double *b, double *gradn)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
gradn[i] -= alpha*gradc[i]*a[i] / (b[i]* b[i]);
}
}