#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vsquare(const double *a, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = a[i] * a[i];
}