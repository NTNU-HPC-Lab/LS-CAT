#include "hip/hip_runtime.h"
#include "includes.h"
// filename: gaxpy2.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "gaxpy2"
{
}
__global__ void vdivbang(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
c[i] = alpha*a[i] / b[i];
}
}