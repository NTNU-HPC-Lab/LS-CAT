#include "hip/hip_runtime.h"
#include "includes.h"
// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmultx"
{
}
__global__ void vmultx(const double alpha, const double *a, const double *b, double *c)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
c[i] = alpha*a[i] * b[i];
}