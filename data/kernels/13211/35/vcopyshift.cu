#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vcopyshift(const int n, const int shift, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
b[i+shift] = a[i];
}
}