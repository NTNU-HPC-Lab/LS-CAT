#include "hip/hip_runtime.h"
#include "includes.h"
// B=diagm(A)

extern "C"
{
}
__global__ void diagm_kernel(const int lengthA, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<lengthA)
{
b[i+i*lengthA] = a[i];
}
}