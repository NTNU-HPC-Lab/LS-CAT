#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vsign(const int n, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
if (a[i]<0)
{b[i]=-1.0;}
else
{if (a[i]>0)
{b[i]=1.0;}
else
{b[i]=0.0;}
}
}
}