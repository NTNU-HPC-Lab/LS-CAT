#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void kinklin(const int n, const double gamma, const double *a, double *b)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<n)
{
if (a[i]>0.0)
{b[i] = a[i];}
else
{b[i] = gamma*a[i];}
}
}