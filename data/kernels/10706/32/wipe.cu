#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wipe(int *buffer, int length) {
length >>= 5;
int tid = threadIdx.x;
for(int i = 0; i < length; i++) {
buffer[(i << 5) + tid] = -1;
}
}