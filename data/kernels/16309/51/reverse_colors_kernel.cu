#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverse_colors_kernel(int num_rows, int max_color, int *row_colors)
{
int row_id = blockIdx.x * blockDim.x + threadIdx.x;

for ( ; row_id < num_rows ; row_id += blockDim.x * gridDim.x )
{
int color = row_colors[row_id];

if (color > 0)
{
//1 -> max_color
//max_color -> 1
color = max_color - color + 1;
}

row_colors[row_id] = color;
}
}