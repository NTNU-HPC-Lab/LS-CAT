#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
// Calculate the row index of the Pd element and M
int Row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
// Calculate the column idenx of Pd and N
int Col = blockIdx.x*BLOCK_SIZE + threadIdx.x;
float Pvalue = 0;
// each thread computes one element of the block sub-matrix
for (int k = 0; k < Width; ++k)
Pvalue += Md[Row*Width+k] * Nd[k*Width+Col];
Pd[Row*Width+Col] = Pvalue;
}