#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_Comput (int *h, int N, int T) {

// Array loaded with global thread ID that acesses that location

int col = threadIdx.x + blockDim.x * blockIdx.x;
int row = threadIdx.y + blockDim.y * blockIdx.y;

int threadID = col + row * N;
int index = row + col * N;		// sequentially down each row

for (int t = 0; t < T; t++)		// loop to repeat to reduce other time effects
h[index] = threadID;  		// load array with flattened global thread ID
}