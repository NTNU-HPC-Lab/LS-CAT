#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuSmMM( float *Ad , float *Bd , float *Cd , int dimention )
{

//Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
__shared__ float Ads [tilewidth][tilewidth] ;
__shared__ float Bds [tilewidth][tilewidth] ;
// calculate thread id
unsigned int col = tilewidth*blockIdx.x + threadIdx.x ;
unsigned int row = tilewidth*blockIdx.y + threadIdx.y ;
for (int m = 0 ; m<dimention/tilewidth ; m++ ) // m indicate number of phase
{
Ads[threadIdx.y][threadIdx.x] =  Ad[row*dimention + (m*tilewidth + threadIdx.x)]  ;
Bds[threadIdx.y][threadIdx.x] =  Bd[ ( m*tilewidth + threadIdx.y) * dimention + col] ;
__syncthreads() ; // for syncronizeing the threads
// Do for tile
for ( int k1 = 0; k1<tilewidth ; k1++ )
Cd[row*dimention + col]+= Ads[threadIdx.x][k1] * Bds[k1][threadIdx.y] ;
__syncthreads() ; // for syncronizeing the threads

}
}