#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuMM(float *A, float *B, float *C, int N)
{
// Matrix multiplication for NxN matrices C=A*B
// Each thread computes a single element of C
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

float sum = 0.f;
for (int n = 0; n < N; ++n)
sum += A[row*N+n]*B[n*N+col];

C[row*N+col] = sum;
}