#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrixmult(int *gpu_a, int *gpu_b, int *gpu_c, int N) {

int k, sum = 0;
int col = threadIdx.x + blockDim.x * blockIdx.x;
int row = threadIdx.y + blockDim.y * blockIdx.y;

if(col < N && row < N) {
for(k = 0; k < N; k++)
sum += gpu_a[row * N + k] * gpu_b[k * N + col];
gpu_c[row * N + col] = sum;
}
}