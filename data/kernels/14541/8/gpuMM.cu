#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuMM(float *A, float *B, float *C, int N)
{
// Matrix multiplication for NxN matrices C=A*B
// Each thread computes a single element of C
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;

float sum = 0.0;
for (int n = 0; n < N; ++n)
sum += A[row*N+n]*B[n*N+col];

C[row*N+col] = sum;

//	if(row%50 ==5)
//		printf("%f \t %f \t %f\n",A[row*N+col], B[row*N+col], C[row*N+col]);

}