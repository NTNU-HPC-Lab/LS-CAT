#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "check_collisions.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float x1_robot = 1;
float y1_robot = 1;
float x2_robot = 1;
float y2_robot = 1;
float *x1_obs = NULL;
hipMalloc(&x1_obs, XSIZE*YSIZE);
float *y1_obs = NULL;
hipMalloc(&y1_obs, XSIZE*YSIZE);
float *x2_obs = NULL;
hipMalloc(&x2_obs, XSIZE*YSIZE);
float *y2_obs = NULL;
hipMalloc(&y2_obs, XSIZE*YSIZE);
bool *collisions = NULL;
hipMalloc(&collisions, XSIZE*YSIZE);
int *indexes = NULL;
hipMalloc(&indexes, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
check_collisions<<<gridBlock,threadBlock>>>(x1_robot,y1_robot,x2_robot,y2_robot,x1_obs,y1_obs,x2_obs,y2_obs,collisions,indexes);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
check_collisions<<<gridBlock,threadBlock>>>(x1_robot,y1_robot,x2_robot,y2_robot,x1_obs,y1_obs,x2_obs,y2_obs,collisions,indexes);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
check_collisions<<<gridBlock,threadBlock>>>(x1_robot,y1_robot,x2_robot,y2_robot,x1_obs,y1_obs,x2_obs,y2_obs,collisions,indexes);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}