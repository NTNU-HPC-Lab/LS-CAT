#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float logit1(const float x) {
return expf(x) / (1. + expf(x));
}
__global__ void logit(float* y, const float* x, int leng) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < leng) {
y[ i ] = logit1(x[ i ]);
}
}