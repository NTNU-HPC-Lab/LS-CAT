#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_double(double* x, int leng) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < leng) {
printf("%lf,", x[ i ]);
}
}