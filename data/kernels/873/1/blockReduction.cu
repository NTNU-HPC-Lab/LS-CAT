#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blockReduction(double* dN_pTdpTdphidy_d, int final_spectrum_size, int blocks_ker1)
{
long idx = threadIdx.x + blockDim.x * blockIdx.x;
if (idx < final_spectrum_size)
{
if (blocks_ker1 == 1) return; //Probably will never happen, but best to be careful
//Need to start at i=1, since adding everything to i=0
for (int i = 1; i < blocks_ker1; i++)
{
dN_pTdpTdphidy_d[idx] += dN_pTdpTdphidy_d[idx + i * final_spectrum_size];
if (isnan(dN_pTdpTdphidy_d[idx])) printf("found dN_pTdpTdphidy_d nan \n");
}
}
}