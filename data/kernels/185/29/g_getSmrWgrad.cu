#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSmrWgrad(float* wgrad, float* weight, float lambda, int len, int batch)
{
for(int i = 0; i < len; i += blockDim.x)
{
int id = i + threadIdx.x;
if(id < len)
{
wgrad[id] = lambda * weight[id] + wgrad[id] / batch;
}
}
}