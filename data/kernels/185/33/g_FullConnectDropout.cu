#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_FullConnectDropout(float * outputs, float * drop, int len)
{
for(int i = 0; i < len; i += blockDim.x * gridDim.x)
{
int id = i + blockIdx.x * blockDim.x + threadIdx.x;
if(id < len)
{
outputs[id] = outputs[id] * drop[id];
}
}
}