#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_feedforward( float* _inputs, float* _w, float* _b, float* _outputs, int rows, int cols, int channels)
{
int row     = blockIdx.x;
int channel = blockIdx.y;

int skip = channel * rows * cols + row * cols;
float* inputs = _inputs + skip;
float* outputs= _outputs+ skip;
// 	if(threadIdx.x == 0)
// 		sprintf(logStr, "block(%d %d) skip = %d\n", blockIdx.x, blockIdx.y, skip);
float* w = _w + channel * cols;
float* b = _b + channel * cols;

for(int i = 0; i < cols; i += blockDim.x){
int id = i + threadIdx.x;
if(id < cols){
outputs[id] = inputs[id] * w[id] + b[id];
}
}
}