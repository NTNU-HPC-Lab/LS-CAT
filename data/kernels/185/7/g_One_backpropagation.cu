#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_backpropagation( float* _curDelta, float* _w, float* _nextDelta, int rows, int cols, int channels)
{
int row     = blockIdx.x;
int channel = blockIdx.y;

int skip = channel * rows * cols + row * cols;
float* curDelta = _curDelta + skip;
float* nextDelta= _nextDelta+ skip;

float* w = _w + channel * cols;

for(int i = 0; i < cols; i += blockDim.x){
int id = i + threadIdx.x;
if(id < cols){
nextDelta[id] = curDelta[id] * w[id];
}
}
}