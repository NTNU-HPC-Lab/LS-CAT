#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols)
{
int bid = blockIdx.x;
extern __shared__ float _share[];
float * _max = _share;
float * _sum = _share + blockDim.x;
float* sp = softMaxP + bid * cols;
_sum[threadIdx.x] = 0.0;
_max[threadIdx.x] = -100000000.0;
for(int tid = 0; tid < cols; tid += blockDim.x){
int id = tid + threadIdx.x;
if(id < cols){
sp[id] += b[id];
_max[threadIdx.x] = max(_max[threadIdx.x], sp[id]);
}
}
__syncthreads();
int len = blockDim.x;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
if(_max[threadIdx.x] < _max[threadIdx.x + skip])
{
_max[threadIdx.x] = _max[threadIdx.x + skip];
}
}
len = (len + 1) >> 1;
}
__syncthreads();
for(int tid = 0; tid < cols; tid += blockDim.x){
int id = tid + threadIdx.x;
if(id < cols){
sp[id] -= _max[0];
sp[id] = __expf(sp[id]);
_sum[threadIdx.x] += sp[id];
}
}
__syncthreads();
len = blockDim.x;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
_sum[threadIdx.x] += _sum[threadIdx.x + skip];
}
len = (len + 1) >> 1;
}
__syncthreads();
for(int tid = 0; tid < cols; tid += blockDim.x){
int id = tid + threadIdx.x;
if(id < cols){
sp[id] /= _sum[0];
}
}
}