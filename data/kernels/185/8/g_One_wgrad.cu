#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_wgrad( float* _inputs, float* _curDelta, float* _wgradTmp, int rows, int cols, int channels)
{
int row     = blockIdx.x;
int channel = blockIdx.y;

int skip = channel * rows * cols + row * cols;
float* inputs   = _inputs   + skip;
float* curDelta = _curDelta + skip;
float* wgradTmp = _wgradTmp + skip;

for(int i = 0; i < cols; i += blockDim.x){
int id = i + threadIdx.x;
if(id < cols){
wgradTmp[id] = inputs[id] * curDelta[id];
}
}
}