#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_wgrad_Add( float* _WgradTmp, float* Wgrad, float* w, int rows, int cols, int channels, float lambda)
{
extern __shared__ float _sum[];
int channel = blockIdx.x;
int col     = blockIdx.y;
int tid     = threadIdx.x;
_sum[tid] = 0;
__syncthreads();


for(int i = 0; i < rows; i += blockDim.x){
int row = i + threadIdx.x;
if(row < rows){
_sum[threadIdx.x] += _WgradTmp[channel * rows * cols + row * cols + col];
}
}
__syncthreads();

int len = rows;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(tid < (len >> 1))
{
_sum[tid] += _sum[tid + skip];
}
len = (len + 1) >> 1;
}
__syncthreads();
if(tid == 0)
{
Wgrad[channel * cols + col] = _sum[0] / rows + w[channel * cols + col] * lambda;
}
}