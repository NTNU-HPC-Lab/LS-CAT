#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_FullConnectWgrad(float* wgrad, float* w, int len, float lambda, int batch)
{
for(int i = 0; i < len; i += blockDim.x * gridDim.x)
{
int id = i + blockDim.x * blockIdx.x + threadIdx.x;
if(id < len)
{
if(fabs(lambda) < 1e-10)
wgrad[id] = wgrad[id] / batch /** dropM[id]*/;
else
wgrad[id] = (wgrad[id] / batch + lambda * w[id]) /** dropM[id]*/;
}
}
}