#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSoftMaxDelta(float* softMaxDelta, float* softMaxP, float* groudTruth, int len)
{
for(int i = 0; i < len; i += blockDim.x)
{
int id = i + threadIdx.x;
if(id < len)
{
softMaxDelta[id] = softMaxP[id] - groudTruth[id];
}
}
}