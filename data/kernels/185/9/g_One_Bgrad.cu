#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_Bgrad(float* _delta, float* bgrad, int rows, int cols, int channels)
{
extern __shared__ float _sum[];
int channel = blockIdx.x;
int col     = blockIdx.y;
int row     = threadIdx.x;
float delta = _delta[channel * rows * cols + row * cols + col];
_sum[row] = delta;
__syncthreads();

int len = rows;
while(len != 1)
{
__syncthreads();
int skip = (len + 1) >> 1;
if(threadIdx.x < (len >> 1))
{
_sum[threadIdx.x] += _sum[threadIdx.x + skip];
}
len = (len + 1) >> 1;
}
__syncthreads();
if(threadIdx.x == 0)
{
bgrad[channel * cols + col] = _sum[0] / rows;
}
}