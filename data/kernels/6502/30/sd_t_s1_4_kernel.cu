#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_4_kernel(int h1d,int h2d,int h3d,int p4d,int p5d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int p5ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p5ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
int h1,h2,h3,p6,p5;
__shared__ double t2_shm[T1*2*Tcomm];

for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
if(i<h1d*p4d)
t2_shm[i] = t2_d[i];
int rest_x=blockIdx.x;
int thread_x = T2*T1 * rest_x + threadIdx.x;
rest_x = thread_x;
__syncthreads();
/* the following computation may need to happen inside the loop */
for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
{
rest_x += i;
h3=rest_x%h3d;
rest_x=rest_x/h3d;
h2=rest_x%h2d;
rest_x=rest_x/h2d;
p6=rest_x%p6d;
rest_x=rest_x/p6d;
p5=rest_x%p5d;

if((thread_x+i)<total_x)
for(h1=0;h1<h1d;h1++)
for(p4=0;p4<p4d;p4++)
{
t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p5*p5ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2+p5*p5ld_v2];
}
}
__syncthreads();
}