#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_2_kernel(size_t h1d,size_t h2d,size_t h3d,size_t p4d,size_t p6d,size_t p4ld_t2,size_t h1ld_t2,size_t h3ld_v2,size_t h2ld_v2,size_t p6ld_v2,size_t h3ld_t3,size_t h2ld_t3,size_t h1ld_t3,size_t p6ld_t3,size_t p4ld_t3,double *t2_d, double *v2_d,size_t p4, size_t total_x, double* t3d) {
size_t h1,h2,h3,p6;
__shared__ double t2_shm[T1*4*Tcomm];

for(size_t i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
if(i<h1d*p4d)
t2_shm[i] = t2_d[i];
size_t rest_x=blockIdx.x;
size_t thread_x = T2*T1 * rest_x + threadIdx.x;
rest_x = thread_x;
__syncthreads();
/* the following computation may need to happen inside the loop */
for(size_t i=0;i<total_x;i+=gridDim.x*blockDim.x)
{
rest_x += i;
h3=rest_x%h3d;
rest_x=rest_x/h3d;
h2=rest_x%h2d;
rest_x=rest_x/h2d;
p6=rest_x%p6d;

if((thread_x+i)<total_x)
for(h1=0;h1<h1d;h1++)
for(p4=0;p4<p4d;p4++)
{
t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2];
}
}
__syncthreads();
}