#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "sd_t_s1_2_kernel.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
size_t h1d = 1;
size_t h2d = 1;
size_t h3d = 1;
size_t p4d = 1;
size_t p6d = 1;
size_t p4ld_t2 = 1;
size_t h1ld_t2 = 1;
size_t h3ld_v2 = 1;
size_t h2ld_v2 = 1;
size_t p6ld_v2 = 1;
size_t h3ld_t3 = 1;
size_t h2ld_t3 = 1;
size_t h1ld_t3 = 1;
size_t p6ld_t3 = 1;
size_t p4ld_t3 = 1;
double *t2_d = NULL;
hipMalloc(&t2_d, XSIZE*YSIZE);
double *v2_d = NULL;
hipMalloc(&v2_d, XSIZE*YSIZE);
size_t p4 = 1;
size_t total_x = 1;
double *t3d = NULL;
hipMalloc(&t3d, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sd_t_s1_2_kernel<<<gridBlock,threadBlock>>>(h1d,h2d,h3d,p4d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,p4,total_x,t3d);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
sd_t_s1_2_kernel<<<gridBlock,threadBlock>>>(h1d,h2d,h3d,p4d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,p4,total_x,t3d);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
sd_t_s1_2_kernel<<<gridBlock,threadBlock>>>(h1d,h2d,h3d,p4d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,p4,total_x,t3d);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}