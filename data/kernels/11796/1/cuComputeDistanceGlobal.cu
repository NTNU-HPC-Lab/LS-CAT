#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuComputeDistanceGlobal(float* A, int wA, int pA, float* B, int wB, int pB, int dim, float* AB){

// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

// Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
__shared__ int begin_A;
__shared__ int begin_B;
__shared__ int step_A;
__shared__ int step_B;
__shared__ int end_A;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Other variables
float tmp;
float ssd = 0;

// Loop parameters
begin_A = BLOCK_DIM * blockIdx.y;
begin_B = BLOCK_DIM * blockIdx.x;
step_A = BLOCK_DIM * pA;
step_B = BLOCK_DIM * pB;
end_A = begin_A + (dim - 1) * pA;

// Conditions
int cond0 = (begin_A + tx < wA); // used to write in shared memory
int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

// Load the matrices from device memory to shared memory; each thread loads one element of each matrix
if (a / pA + ty < dim){
shared_A[ty][tx] = (cond0) ? A[a + pA * ty + tx] : 0;
shared_B[ty][tx] = (cond1) ? B[b + pB * ty + tx] : 0;
}
else{
shared_A[ty][tx] = 0;
shared_B[ty][tx] = 0;
}

// Synchronize to make sure the matrices are loaded
__syncthreads();

// Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
if (cond2 && cond1){
for (int k = 0; k < BLOCK_DIM; ++k){
tmp = shared_A[k][ty] - shared_B[k][tx];
ssd += tmp*tmp;
}
}

// Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
__syncthreads();
}

// Write the block sub-matrix to device memory; each thread writes one element
if (cond2 && cond1)
AB[(begin_A + ty) * pB + begin_B + tx] = ssd;
}