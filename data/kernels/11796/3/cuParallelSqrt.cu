#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
if (xIndex<width && yIndex<k)
dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}