#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void updateU(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, const double *d_u1Temp, const double *d_u2Temp, const double *d_u3Temp) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < nbrOfGrids; i += stride) {
if ((i > 0) && (i < nbrOfGrids - 1)) {
d_u1[i] = d_u1Temp[i];
d_u2[i] = d_u2Temp[i];
d_u3[i] = d_u3Temp[i];
}
}
}
__device__ void step(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < nbrOfGrids; i += stride) {
if ((i > 0) && (i < nbrOfGrids - 1)) {
d_u1Temp[i] = d_u1[i] - *d_tau / *d_h * (d_f1[i] - d_f1[i - 1]);
d_u2Temp[i] = d_u2[i] - *d_tau / *d_h * (d_f2[i] - d_f2[i - 1]);
d_u3Temp[i] = d_u3[i] - *d_tau / *d_h * (d_f3[i] - d_f3[i - 1]);
}
}
}
__device__ void halfStep(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < nbrOfGrids; i += stride) {
if ((i > 0) && (i < nbrOfGrids - 1)) {
d_u1Temp[i] = (d_u1[i + 1] + d_u1[i]) / 2 - *d_tau / 2 / *d_h * (d_f1[i + 1] - d_f1[i]);
d_u2Temp[i] = (d_u2[i + 1] + d_u2[i]) / 2 - *d_tau / 2 / *d_h * (d_f2[i + 1] - d_f2[i]);
d_u3Temp[i] = (d_u3[i + 1] + d_u3[i]) / 2 - *d_tau / 2 / *d_h * (d_f3[i + 1] - d_f3[i]);
}
}
}
__device__ void updateFlux(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_f1, double *d_f2, double *d_f3, const double *d_gama) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
double rho, m, e, p;
for (int i = index; i < nbrOfGrids; i += stride) {
rho = d_u1[i];
m = d_u2[i];
e = d_u3[i];
p = (*d_gama - 1) * (e - m * m / rho / 2);
d_f1[i] = m;
d_f2[i] = m * m / rho + p;
d_f3[i] = m / rho * (e + p);
}
}
__device__ void d_boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
d_u1[0] = d_u1[1];
d_u2[0] = -d_u2[1];
d_u3[0] = d_u3[1];
d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}
__global__	void laxWendroffStep(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama) {
updateFlux(nbrOfGrids, d_u1, d_u2, d_u3, d_f1, d_f2, d_f3, d_gama);
halfStep(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
d_boundaryCondition(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp);
updateFlux(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_gama);
step(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
updateU(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp);
d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
}