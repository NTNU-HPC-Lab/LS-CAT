#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "laxWendroffStep.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int nbrOfGrids = 1;
double *d_u1 = NULL;
hipMalloc(&d_u1, XSIZE*YSIZE);
double *d_u2 = NULL;
hipMalloc(&d_u2, XSIZE*YSIZE);
double *d_u3 = NULL;
hipMalloc(&d_u3, XSIZE*YSIZE);
double *d_u1Temp = NULL;
hipMalloc(&d_u1Temp, XSIZE*YSIZE);
double *d_u2Temp = NULL;
hipMalloc(&d_u2Temp, XSIZE*YSIZE);
double *d_u3Temp = NULL;
hipMalloc(&d_u3Temp, XSIZE*YSIZE);
double *d_f1 = NULL;
hipMalloc(&d_f1, XSIZE*YSIZE);
double *d_f2 = NULL;
hipMalloc(&d_f2, XSIZE*YSIZE);
double *d_f3 = NULL;
hipMalloc(&d_f3, XSIZE*YSIZE);
const double *d_tau = NULL;
hipMalloc(&d_tau, XSIZE*YSIZE);
const double *d_h = NULL;
hipMalloc(&d_h, XSIZE*YSIZE);
const double *d_gama = NULL;
hipMalloc(&d_gama, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
laxWendroffStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_u1Temp,d_u2Temp,d_u3Temp,d_f1,d_f2,d_f3,d_tau,d_h,d_gama);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
laxWendroffStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_u1Temp,d_u2Temp,d_u3Temp,d_f1,d_f2,d_f3,d_tau,d_h,d_gama);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
laxWendroffStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_u1Temp,d_u2Temp,d_u3Temp,d_f1,d_f2,d_f3,d_tau,d_h,d_gama);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}