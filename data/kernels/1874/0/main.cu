#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "initDeviceMemory.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int nbrOfGrids = 1;
double *d_u1 = NULL;
hipMalloc(&d_u1, XSIZE*YSIZE);
double *d_u2 = NULL;
hipMalloc(&d_u2, XSIZE*YSIZE);
double *d_u3 = NULL;
hipMalloc(&d_u3, XSIZE*YSIZE);
double *d_vol = NULL;
hipMalloc(&d_vol, XSIZE*YSIZE);
double *d_h = NULL;
hipMalloc(&d_h, XSIZE*YSIZE);
double *d_length = NULL;
hipMalloc(&d_length, XSIZE*YSIZE);
double *d_gama = NULL;
hipMalloc(&d_gama, XSIZE*YSIZE);
double *d_cfl = NULL;
hipMalloc(&d_cfl, XSIZE*YSIZE);
double *d_nu = NULL;
hipMalloc(&d_nu, XSIZE*YSIZE);
double *d_tau = NULL;
hipMalloc(&d_tau, XSIZE*YSIZE);
double *d_cMax = NULL;
hipMalloc(&d_cMax, XSIZE*YSIZE);
double *d_t = NULL;
hipMalloc(&d_t, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
initDeviceMemory<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_h,d_length,d_gama,d_cfl,d_nu,d_tau,d_cMax,d_t);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
initDeviceMemory<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_h,d_length,d_gama,d_cfl,d_nu,d_tau,d_cMax,d_t);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
initDeviceMemory<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_h,d_length,d_gama,d_cfl,d_nu,d_tau,d_cMax,d_t);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}