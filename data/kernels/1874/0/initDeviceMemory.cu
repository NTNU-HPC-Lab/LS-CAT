#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax)
{
*d_cMax = 0; int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
double ro, p, u;
__shared__ double c;
for (int i = index; i < nbrOfGrids; i += stride){
if (d_u1[i] == 0)
continue;
ro = d_u1[i];
u = d_u2[i] / ro;
p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
c = sqrt(*d_gama * abs(p) / ro);
if (*d_cMax < c + abs(u))
*d_cMax = c + abs(u);
}
}
__global__ void initDeviceMemory(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, double *d_vol, double *d_h, double *d_length, double *d_gama, double *d_cfl, double *d_nu, double *d_tau, double *d_cMax, double *d_t) {
*d_t = 0;								// time
*d_length = 1;							// length of shock tube
*d_gama = 1.4;							// ratio of specific heats
*d_cfl = 0.9;							// Courant-Friedrichs-Lewy number
*d_nu = 0.0;							// artificial viscosity coefficient
*d_h = *d_length / (nbrOfGrids - 1);	// space grid size
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
for(int i = index; i < nbrOfGrids; i+= stride){
double e, ro, p, u = 0;
if (i < nbrOfGrids){
if (i >= int(nbrOfGrids / 2)) { ro = 0.125, p = 0.1; }
else { ro = 1, p = 1; }
e = p / (*d_gama - 1) + ro * u * u / 2;
d_u1[i] = ro;
d_u2[i] = ro * u;
d_u3[i] = e;
d_u3[i] = e;
d_vol[i] = 1;
}
}
updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
*d_tau = (*d_cfl) * (*d_h) / (*d_cMax);    // initial time grid size, It will be modified to tMax if this > tMax
}