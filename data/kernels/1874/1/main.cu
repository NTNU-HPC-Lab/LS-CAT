#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "updateTau.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int nbrOfGrids = 1;
const double *d_u1 = NULL;
hipMalloc(&d_u1, XSIZE*YSIZE);
const double *d_u2 = NULL;
hipMalloc(&d_u2, XSIZE*YSIZE);
const double *d_u3 = NULL;
hipMalloc(&d_u3, XSIZE*YSIZE);
const double *d_gama = NULL;
hipMalloc(&d_gama, XSIZE*YSIZE);
double *d_cMax = NULL;
hipMalloc(&d_cMax, XSIZE*YSIZE);
const double *d_h = NULL;
hipMalloc(&d_h, XSIZE*YSIZE);
const double *d_cfl = NULL;
hipMalloc(&d_cfl, XSIZE*YSIZE);
double *d_tau = NULL;
hipMalloc(&d_tau, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
updateTau<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_gama,d_cMax,d_h,d_cfl,d_tau);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
updateTau<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_gama,d_cMax,d_h,d_cfl,d_tau);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
updateTau<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_gama,d_cMax,d_h,d_cfl,d_tau);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}