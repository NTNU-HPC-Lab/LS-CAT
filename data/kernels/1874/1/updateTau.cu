#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax)
{
*d_cMax = 0; int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
double ro, p, u;
__shared__ double c;
for (int i = index; i < nbrOfGrids; i += stride){
if (d_u1[i] == 0)
continue;
ro = d_u1[i];
u = d_u2[i] / ro;
p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
c = sqrt(*d_gama * abs(p) / ro);
if (*d_cMax < c + abs(u))
*d_cMax = c + abs(u);
}
}
__global__ void updateTau(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax, const double *d_h, const double *d_cfl, double *d_tau) {
updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
*d_tau = *d_cfl * *d_h / *d_cMax;
}