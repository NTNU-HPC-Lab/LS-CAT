#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "RoeStep.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const int nbrOfGrids = 1;
double *d_u1 = NULL;
hipMalloc(&d_u1, XSIZE*YSIZE);
double *d_u2 = NULL;
hipMalloc(&d_u2, XSIZE*YSIZE);
double *d_u3 = NULL;
hipMalloc(&d_u3, XSIZE*YSIZE);
const double *d_vol = NULL;
hipMalloc(&d_vol, XSIZE*YSIZE);
double *d_f1 = NULL;
hipMalloc(&d_f1, XSIZE*YSIZE);
double *d_f2 = NULL;
hipMalloc(&d_f2, XSIZE*YSIZE);
double *d_f3 = NULL;
hipMalloc(&d_f3, XSIZE*YSIZE);
const double *d_tau = NULL;
hipMalloc(&d_tau, XSIZE*YSIZE);
const double *d_h = NULL;
hipMalloc(&d_h, XSIZE*YSIZE);
const double *d_gama = NULL;
hipMalloc(&d_gama, XSIZE*YSIZE);
double *w1 = NULL;
hipMalloc(&w1, XSIZE*YSIZE);
double *w2 = NULL;
hipMalloc(&w2, XSIZE*YSIZE);
double *w3 = NULL;
hipMalloc(&w3, XSIZE*YSIZE);
double *w4 = NULL;
hipMalloc(&w4, XSIZE*YSIZE);
double *fc1 = NULL;
hipMalloc(&fc1, XSIZE*YSIZE);
double *fc2 = NULL;
hipMalloc(&fc2, XSIZE*YSIZE);
double *fc3 = NULL;
hipMalloc(&fc3, XSIZE*YSIZE);
double *fr1 = NULL;
hipMalloc(&fr1, XSIZE*YSIZE);
double *fr2 = NULL;
hipMalloc(&fr2, XSIZE*YSIZE);
double *fr3 = NULL;
hipMalloc(&fr3, XSIZE*YSIZE);
double *fl1 = NULL;
hipMalloc(&fl1, XSIZE*YSIZE);
double *fl2 = NULL;
hipMalloc(&fl2, XSIZE*YSIZE);
double *fl3 = NULL;
hipMalloc(&fl3, XSIZE*YSIZE);
double *fludif1 = NULL;
hipMalloc(&fludif1, XSIZE*YSIZE);
double *fludif2 = NULL;
hipMalloc(&fludif2, XSIZE*YSIZE);
double *fludif3 = NULL;
hipMalloc(&fludif3, XSIZE*YSIZE);
double *rsumr = NULL;
hipMalloc(&rsumr, XSIZE*YSIZE);
double *utilde = NULL;
hipMalloc(&utilde, XSIZE*YSIZE);
double *htilde = NULL;
hipMalloc(&htilde, XSIZE*YSIZE);
double *uvdif = NULL;
hipMalloc(&uvdif, XSIZE*YSIZE);
double *absvt = NULL;
hipMalloc(&absvt, XSIZE*YSIZE);
double *ssc = NULL;
hipMalloc(&ssc, XSIZE*YSIZE);
double *vsc = NULL;
hipMalloc(&vsc, XSIZE*YSIZE);
double *eiglam1 = NULL;
hipMalloc(&eiglam1, XSIZE*YSIZE);
double *eiglam2 = NULL;
hipMalloc(&eiglam2, XSIZE*YSIZE);
double *eiglam3 = NULL;
hipMalloc(&eiglam3, XSIZE*YSIZE);
double *sgn1 = NULL;
hipMalloc(&sgn1, XSIZE*YSIZE);
double *sgn2 = NULL;
hipMalloc(&sgn2, XSIZE*YSIZE);
double *sgn3 = NULL;
hipMalloc(&sgn3, XSIZE*YSIZE);
int *isb1 = NULL;
hipMalloc(&isb1, XSIZE*YSIZE);
int *isb2 = NULL;
hipMalloc(&isb2, XSIZE*YSIZE);
int *isb3 = NULL;
hipMalloc(&isb3, XSIZE*YSIZE);
double *a1 = NULL;
hipMalloc(&a1, XSIZE*YSIZE);
double *a2 = NULL;
hipMalloc(&a2, XSIZE*YSIZE);
double *a3 = NULL;
hipMalloc(&a3, XSIZE*YSIZE);
double *ac11 = NULL;
hipMalloc(&ac11, XSIZE*YSIZE);
double *ac12 = NULL;
hipMalloc(&ac12, XSIZE*YSIZE);
double *ac13 = NULL;
hipMalloc(&ac13, XSIZE*YSIZE);
double *ac21 = NULL;
hipMalloc(&ac21, XSIZE*YSIZE);
double *ac22 = NULL;
hipMalloc(&ac22, XSIZE*YSIZE);
double *ac23 = NULL;
hipMalloc(&ac23, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
RoeStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_f1,d_f2,d_f3,d_tau,d_h,d_gama,w1,w2,w3,w4,fc1,fc2,fc3,fr1,fr2,fr3,fl1,fl2,fl3,fludif1,fludif2,fludif3,rsumr,utilde,htilde,uvdif,absvt,ssc,vsc,eiglam1,eiglam2,eiglam3,sgn1,sgn2,sgn3,isb1,isb2,isb3,a1,a2,a3,ac11,ac12,ac13,ac21,ac22,ac23);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
RoeStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_f1,d_f2,d_f3,d_tau,d_h,d_gama,w1,w2,w3,w4,fc1,fc2,fc3,fr1,fr2,fr3,fl1,fl2,fl3,fludif1,fludif2,fludif3,rsumr,utilde,htilde,uvdif,absvt,ssc,vsc,eiglam1,eiglam2,eiglam3,sgn1,sgn2,sgn3,isb1,isb2,isb3,a1,a2,a3,ac11,ac12,ac13,ac21,ac22,ac23);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
RoeStep<<<gridBlock,threadBlock>>>(nbrOfGrids,d_u1,d_u2,d_u3,d_vol,d_f1,d_f2,d_f3,d_tau,d_h,d_gama,w1,w2,w3,w4,fc1,fc2,fc3,fr1,fr2,fr3,fl1,fl2,fl3,fludif1,fludif2,fludif3,rsumr,utilde,htilde,uvdif,absvt,ssc,vsc,eiglam1,eiglam2,eiglam3,sgn1,sgn2,sgn3,isb1,isb2,isb3,a1,a2,a3,ac11,ac12,ac13,ac21,ac22,ac23);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}