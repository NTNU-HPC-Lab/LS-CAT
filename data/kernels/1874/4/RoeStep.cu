#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void d_boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
d_u1[0] = d_u1[1];
d_u2[0] = -d_u2[1];
d_u3[0] = d_u3[1];
d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}
__global__	void RoeStep(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, const double *d_vol, double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama, double *w1,double *w2,double *w3,double *w4, double *fc1,double *fc2,double *fc3, double *fr1,double *fr2,double *fr3, double *fl1,double *fl2,double *fl3, double *fludif1,double *fludif2,double *fludif3, double *rsumr, double *utilde, double *htilde, double *uvdif, double *absvt, double *ssc, double *vsc, double *eiglam1,double *eiglam2,double *eiglam3, double *sgn1,double *sgn2,double *sgn3, int *isb1,int *isb2,int *isb3, double *a1,double *a2,double *a3, double *ac11,double *ac12,double *ac13, double *ac21,double *ac22,double *ac23) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x;
for (int i = index; i < nbrOfGrids; i += stride) {

// find parameter vector w
{
w1[i] = sqrt(d_vol[i] * d_u1[i]);
w2[i] = w1[i] * d_u2[i] / d_u1[i];
w4[i] = (*d_gama - 1) * (d_u3[i] - 0.5 * d_u2[i] * d_u2[i] / d_u1[i]);
w3[i] = w1[i] * (d_u3[i] + w4[i]) / d_u1[i];
}

// calculate the fluxes at the cell center
{
fc1[i] = w1[i] * w2[i];
fc2[i] = w2[i] * w2[i] + d_vol[i] * w4[i];
fc3[i] = w2[i] * w3[i];
}

__syncthreads(); // because of the [i - 1] index below
// calculate the fluxes at the cell walls
if (i > 0) {
fl1[i] = fc1[i - 1]; fr1[i] = fc1[i];
fl2[i] = fc2[i - 1]; fr2[i] = fc2[i];
fl3[i] = fc3[i - 1]; fr3[i] = fc3[i];
}

// calculate the flux differences at the cell walls
if (i > 0) {
fludif1[i] = fr1[i] - fl1[i];
fludif2[i] = fr2[i] - fl2[i];
fludif3[i] = fr3[i] - fl3[i];
}

__syncthreads(); // because of the [i - 1] index below
// calculate the tilded state variables = mean values at the interfaces
if (i > 0) {
rsumr[i] = 1 / (w1[i - 1] + w1[i]);

utilde[i] = (w2[i - 1] + w2[i]) * rsumr[i];
htilde[i] = (w3[i - 1] + w3[i]) * rsumr[i];

absvt[i] = 0.5 * utilde[i] * utilde[i];
uvdif[i] = utilde[i] * fludif2[i];

ssc[i] = (*d_gama - 1) * (htilde[i] - absvt[i]);
if (ssc[i] > 0.0)
vsc[i] = sqrt(ssc[i]);
else {
vsc[i] = sqrt(abs(ssc[i]));
}
}

// calculate the eigenvalues and projection coefficients for each eigenvector
if (i > 0) {
eiglam1[i] = utilde[i] - vsc[i];
eiglam2[i] = utilde[i];
eiglam3[i] = utilde[i] + vsc[i];
sgn1[i] = eiglam1[i] < 0.0 ? -1 : 1;
sgn2[i] = eiglam2[i] < 0.0 ? -1 : 1;
sgn3[i] = eiglam3[i] < 0.0 ? -1 : 1;
a1[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i]
- uvdif[i]) - vsc[i] * (fludif2[i] - utilde[i]
* fludif1[i])) / ssc[i];
a2[i] = (*d_gama - 1) * ((htilde[i] - 2 * absvt[i]) * fludif1[i]
+ uvdif[i] - fludif3[i]) / ssc[i];
a3[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i]
- uvdif[i]) + vsc[i] * (fludif2[i] - utilde[i]
* fludif1[i])) / ssc[i];
}

// divide the projection coefficients by the wave speeds to evade expansion correction
if (i > 0) {
a1[i] /= eiglam1[i] + tiny;
a2[i] /= eiglam2[i] + tiny;
a3[i] /= eiglam3[i] + tiny;
}

// calculate the first order projection coefficients ac1
if (i > 0) {
ac11[i] = -sgn1[i] * a1[i] * eiglam1[i];
ac12[i] = -sgn2[i] * a2[i] * eiglam2[i];
ac13[i] = -sgn3[i] * a3[i] * eiglam3[i];
}

// apply the 'superbee' flux correction to made 2nd order projection coefficients ac2
{
ac21[1] = ac11[1];
ac21[nbrOfGrids - 1] = ac11[nbrOfGrids - 1];
ac22[1] = ac12[1];
ac22[nbrOfGrids - 1] = ac12[nbrOfGrids - 1];
ac23[1] = ac13[1];
ac23[nbrOfGrids - 1] = ac13[nbrOfGrids - 1];


double dtdx = *d_tau / *d_h;
if ((i > 1) && (i < nbrOfGrids - 1)) {
isb1[i] = i - int(sgn1[i]);
ac21[i] = ac11[i] + eiglam1[i] *
((fmax(0.0, fmin(sbpar1 * a1[isb1[i]], fmax(a1[i], fmin(a1[isb1[i]], sbpar2 * a1[i])))) +
fmin(0.0, fmax(sbpar1 * a1[isb1[i]], fmin(a1[i], fmax(a1[isb1[i]], sbpar2 * a1[i]))))) *
(sgn1[i] - dtdx * eiglam1[i]));
isb2[i] = i - int(sgn2[i]);
ac22[i] = ac12[i] + eiglam2[i] *
((fmax(0.0, fmin(sbpar1 * a2[isb2[i]], fmax(a2[i], fmin(a2[isb2[i]], sbpar2 * a2[i])))) +
fmin(0.0, fmax(sbpar1 * a2[isb2[i]], fmin(a2[i], fmax(a2[isb2[i]], sbpar2 * a2[i]))))) *
(sgn2[i] - dtdx * eiglam2[i]));
isb3[i] = i - int(sgn3[i]);
ac23[i] = ac13[i] + eiglam3[i] *
((fmax(0.0, fmin(sbpar1 * a3[isb3[i]], fmax(a3[i], fmin(a3[isb3[i]], sbpar2 * a3[i])))) +
fmin(0.0, fmax(sbpar1 * a3[isb3[i]], fmin(a3[i], fmax(a3[isb3[i]], sbpar2 * a3[i]))))) *
(sgn3[i] - dtdx * eiglam3[i]));
}
}

// calculate the final fluxes
if (i > 0) {
d_f1[i] = 0.5 * (fl1[i] + fr1[i] + ac21[i] + ac22[i] + ac23[i]);
d_f2[i] = 0.5 * (fl2[i] + fr2[i] + eiglam1[i] * ac21[i]
+ eiglam2[i] * ac22[i] + eiglam3[i] * ac23[i]);
d_f3[i] = 0.5 * (fl3[i] + fr3[i] + (htilde[i] - utilde[i] * vsc[i]) * ac21[i]
+ absvt[i] * ac22[i] + (htilde[i] + utilde[i] * vsc[i]) * ac23[i]);
}

__syncthreads(); // because of the [i + 1] index below
// update U
if (i > 0 && i < nbrOfGrids - 1) {
d_u1[i] -= *d_tau / *d_h * (d_f1[i + 1] - d_f1[i]);
d_u2[i] -= *d_tau / *d_h * (d_f2[i + 1] - d_f2[i]);
d_u3[i] -= *d_tau / *d_h * (d_f3[i + 1] - d_f3[i]);
}

d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
}
}