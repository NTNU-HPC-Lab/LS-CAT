#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
d_u1[0] = d_u1[1];
d_u2[0] = -d_u2[1];
d_u3[0] = d_u3[1];
d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}