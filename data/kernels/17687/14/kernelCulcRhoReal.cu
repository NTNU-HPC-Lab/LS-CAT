#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCulcRhoReal(const int N, double *rho, double *q, double *p, const double lambda, const double g)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
double qi = q[i];
double pi = p[i];

rho[i] = 0.5 * qi * qi;
rho[i] += 0.5 * pi * pi;
rho[i] += (lambda / 4.0) * qi * qi * qi * qi;
rho[i] += (g / 6.0)  * qi * qi * qi * qi * qi * qi;
}
}