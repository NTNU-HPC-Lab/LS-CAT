#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGetOmega(const int N, double *omega, double *kSqr, const double sigma2, const double sigma4, const double lambda, const double g)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
omega[i] = sqrt(1 + kSqr[i] + 3 * lambda * sigma2 + 15 * g * sigma4);
}
}