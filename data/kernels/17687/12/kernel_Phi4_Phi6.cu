#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_Phi4_Phi6(const int N, double *t, double *q, const double lambda, const double g)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
t[i] = q[i] * q[i] * q[i] * (lambda + g * q[i] * q[i]);
}
}