#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGetPhi3(const int N, double *T, double *q)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
T[i] = q[i] * q[i] * q[i];
}
}