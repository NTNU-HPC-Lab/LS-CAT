#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelSyncBuf(double *A, double *A0)
{
const int i = threadIdx.x;
const int j = threadIdx.y;
const int k = threadIdx.z;
const int N1 = blockDim.x;
const int N2 = blockDim.y;
const int N3 = blockDim.z;

const int iB = blockIdx.x;
const int jB = blockIdx.y;
const int kB = blockIdx.z;
//const int N1B = gridDim.x;	//just never used
const int N2B = gridDim.y;
const int N3B = gridDim.z;

const int iG = i + iB * N1;
const int jG = j + jB * N2;
const int kG = k + kB * N3;
//const int N1G = N1 * N1B;		//just never used
const int N2G = N2 * N2B;
const int N3G = N3 * N3B;

const int indB = k + N3 * (j + N2 * i);
const int indA = kB + N3B * (jB + N2B * iB);
const int indA0 = kG + N3G * (jG + N2G * iG);

extern __shared__ double B[];
B[indB] = A0[indA0];
__syncthreads();


int numOfElem = N1 * N2 * N3;
int step = 1;
while (numOfElem > 1)
{
if (indB % (2*step) == 0)
{
B[indB] = B[indB] + B[indB + step];
}
__syncthreads();

numOfElem /= 2;
step *= 2;

}

if (indB == 0)
{
A[indA] = B[0] / (N1 * N2 * N3);
}

}