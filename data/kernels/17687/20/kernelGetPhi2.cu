#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGetPhi2(const int N, double *T, double *q)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
T[i] = q[i] * q[i];
}
}