#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelAddMullSqr(const int N, double* S, double* A, double m)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < N)
{
S[i] += m * A[i] * A[i];
}
}