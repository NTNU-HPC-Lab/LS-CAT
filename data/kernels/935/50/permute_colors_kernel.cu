#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void permute_colors_kernel(int num_rows, int *row_colors, int *color_permutation)
{
int row_id = blockIdx.x * blockDim.x + threadIdx.x;

for ( ; row_id < num_rows ; row_id += blockDim.x * gridDim.x )
{
int color = row_colors[row_id];
#if __CUDA_ARCH__ >= 350
color = __ldg(color_permutation + color);
#else
color = color_permutation[color];
#endif
row_colors[row_id] = color;
}
}