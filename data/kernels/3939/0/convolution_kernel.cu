#include "hip/hip_runtime.h"
#include "includes.h"
//Training of the CNN is done using Keras. After training for 10 epochs, the obtained accuracy on the training data set is 99.70 and on the test data set is 99.14.
//This model implements the following layes in order- 2DConvolution---->Maxpooling---->2D Convolution---->Maxpooling---->Fully_connected layer---->Fully_connected layer.
//The image is a 28*28 greyscale image. The specifications of the layers are as follows:
//Layer_0: Convolution: 32 3*3 kernels with no padding and 1 stride.
//Layer_1: Maxpooling: 2*2 filters with with no padding and 1 stride.
//Layer_2: Convolution: 64 3*3 kernels with no padding and 1 stride.
//Layer_3: Maxpooling: 2*2 filters with with no padding and 1 stride.
//Layer_4: Flattening
//Layer_5: Fully connected / dense layer with 1024 output units.
//Layer_6: Dropout (done during training only).
//Layer_7: Fully connected / dense layer with 10 output units.

//All arrays and matrices are designed to be row ordered in this implementation.



//Kernel that does convolution. This convolution is done by each thread identifying that patch or portion of the image that it is responsible for its result and does the multiplication and addition of it's patche's values with the suitable kernel.
//The depth of the output image is the number of kernels.

//Kernel that does maxpooling.

//This kernel implements the fully connected layers.


__global__ void convolution_kernel(int h, int w, int d, double* gpu_in, int k_h, int k_w, int k_d, double* kernel_weights, double* kernel_biases, int num_kernels, int op_h, int op_w, int op_d, double* gpu_out)
{
//Identifying threads by their IDs.
int row = blockDim.y*blockIdx.y + threadIdx.y;
int col = blockDim.x*blockIdx.x + threadIdx.x;
int deep = blockDim.z *blockIdx.z + threadIdx.z;
//Return if thread out of bounds
if (row >= op_h || col >= op_w || deep >= op_d) return;
double out=0.0;
int kernel_pointer = 0;
//Each thread/each output node identifies the corresponding element in the matrix that it is responsible to multiply-add.
for (int depth_pointer = 0; depth_pointer < k_d; depth_pointer++) {
for (int row_pointer = 0; row_pointer < k_h; row_pointer++) {
for (int column_pointer = 0; column_pointer < k_w; column_pointer++) {
out += gpu_in[((row*w + col) + row_pointer * w + column_pointer + h * w*depth_pointer)] * kernel_weights[kernel_pointer + deep * k_h*k_w*k_d];
kernel_pointer++;
}
}
}
//Bias addition and relu activation. One bias is applied to one output image layer, since one bias is applicable to one kernel.
//Relu activation : relu(a)=max(0,a). If the value is less than 0 then it becomes 0, else it is retained.
if (out + kernel_biases[deep] < 0.0)
gpu_out[row*op_w + col + deep * op_h*op_w] = 0.0l;
else
gpu_out[row*op_w + col + deep * op_h*op_w] = out + kernel_biases[deep];

}