#include "hip/hip_runtime.h"
#include "includes.h"
//Training of the CNN is done using Keras. After training for 10 epochs, the obtained accuracy on the training data set is 99.70 and on the test data set is 99.14.
//This model implements the following layes in order- 2DConvolution---->Maxpooling---->2D Convolution---->Maxpooling---->Fully_connected layer---->Fully_connected layer.
//The image is a 28*28 greyscale image. The specifications of the layers are as follows:
//Layer_0: Convolution: 32 3*3 kernels with no padding and 1 stride.
//Layer_1: Maxpooling: 2*2 filters with with no padding and 1 stride.
//Layer_2: Convolution: 64 3*3 kernels with no padding and 1 stride.
//Layer_3: Maxpooling: 2*2 filters with with no padding and 1 stride.
//Layer_4: Flattening
//Layer_5: Fully connected / dense layer with 1024 output units.
//Layer_6: Dropout (done during training only).
//Layer_7: Fully connected / dense layer with 10 output units.

//All arrays and matrices are designed to be row ordered in this implementation.



//Kernel that does convolution. This convolution is done by each thread identifying that patch or portion of the image that it is responsible for its result and does the multiplication and addition of it's patche's values with the suitable kernel.
//The depth of the output image is the number of kernels.

//Kernel that does maxpooling.

//This kernel implements the fully connected layers.


__global__ void dense_kernel(int num_input, int num_output, double* gpu_in, double* weights, double* biases, double* gpu_out, int num_classes)
{
int tid = blockDim.x*blockIdx.x + threadIdx.x;
if (tid >= num_output) return;
double sum = 0.0l;
//The weights are extracted from Keras such that all the weights to one output node appears together, followed by weights to the next node and so on.
//Thus, each output node will be a multiply add of adjacent weight values with the input nodes.
for (int count = 0; count < num_input; count++) {
sum += gpu_in[count] * weights[tid*num_input + count];
}
sum += biases[tid];

//Activation: If the layer is the final layer, then don't do anything, otherwise relu activation max(0,value) is taken.
if ((num_output) != num_classes) {
if (sum < 0.0) {
sum = 0.0l;
}
}
gpu_out[tid] = sum;
}