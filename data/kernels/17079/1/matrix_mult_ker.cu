#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float rowcol_dot(float * matrix_a, float * matrix_b, int row, int col, int N)
{
float val = 0;

for (int k = 0; k < N; k++)
{
val += matrix_a[row*N + k] * matrix_b[col + k*N];
}
return val;
}
__global__ void matrix_mult_ker(float * matrix_a, float * matrix_b, float * output_matrix, int N)
{
int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

output_matrix[col + row * N] = rowcol_dot(matrix_a, matrix_b, row, col, N);
}