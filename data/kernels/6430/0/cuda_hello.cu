#include "hip/hip_runtime.h"
#include "includes.h"

/*
Nvidia Jetson Nano Cuda info
CUDA Device Query (Runtime API) version (CUDART static linking)

Detected 1 CUDA Capable device(s)

Device 0: "NVIDIA Tegra X1"
CUDA Driver Version / Runtime Version          10.0 / 10.0
CUDA Capability Major/Minor version number:    5.3
Total amount of global memory:                 3957 MBytes (4148756480 bytes)
( 1) Multiprocessors, (128) CUDA Cores/MP:     128 CUDA Cores
GPU Max Clock rate:                            922 MHz (0.92 GHz)
Memory Clock rate:                             13 Mhz
Memory Bus Width:                              64-bit
L2 Cache Size:                                 262144 bytes
Maximum Texture Dimension Size (x,y,z)         1D=(65536), 2D=(65536, 65536), 3D=(4096, 4096, 4096)
Maximum Layered 1D Texture Size, (num) layers  1D=(16384), 2048 layers
Maximum Layered 2D Texture Size, (num) layers  2D=(16384, 16384), 2048 layers
Total amount of constant memory:               65536 bytes
Total amount of shared memory per block:       49152 bytes
Total number of registers available per block: 32768
Warp size:                                     32
Maximum number of threads per multiprocessor:  2048
Maximum number of threads per block:           1024
Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
Maximum memory pitch:                          2147483647 bytes
Texture alignment:                             512 bytes
Concurrent copy and kernel execution:          Yes with 1 copy engine(s)
Run time limit on kernels:                     Yes
Integrated GPU sharing Host Memory:            Yes
Support host page-locked memory mapping:       Yes
Alignment requirement for Surfaces:            Yes
Device has ECC support:                        Disabled
Device supports Unified Addressing (UVA):      Yes
Device supports Compute Preemption:            No
Supports Cooperative Kernel Launch:            No
Supports MultiDevice Co-op Kernel Launch:      No
Device PCI Domain ID / Bus ID / location ID:   0 / 0 / 0
Compute Mode:
< Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >
deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 10.0, CUDA Runtime Version = 10.0, NumDevs = 1

Result = PASS
*/
__global__ void cuda_hello(){
printf("Hello World from GPU! %d\n", threadIdx.x*gridDim.x);
}