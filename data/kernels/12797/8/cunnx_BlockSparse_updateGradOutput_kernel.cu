#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_BlockSparse_updateGradOutput_kernel( float *_gradOutput, float* gradOutputScale, const float *gradOutput, const float *output, const float *outputScale, int outputWindowSize, int outputSize)
{
__shared__ float buffer[BLOCKSPARSE_THREADS];
int tx = threadIdx.x;
int i_step = blockDim.x;
int k = blockIdx.x;

float *_gradOutput_k = _gradOutput + k*outputWindowSize*outputSize;
float *gradOutputScale_k = gradOutputScale + k*outputWindowSize;
const float *gradOutput_k = gradOutput + k*outputWindowSize*outputSize;
const float *output_k = output + k*outputWindowSize*outputSize;
const float *outputScale_k = outputScale + k*outputWindowSize;


// get gradients for outputScale (to be backwarded to a Gater)
for (int m=0; m<outputWindowSize; m++)
{
float outputScale = outputScale_k[m];

float *_blockGradOutput = _gradOutput_k + m*outputSize;
const float *blockGradOutput = gradOutput_k + m*outputSize;
const float *blockOutput = output_k + m*outputSize;

buffer[tx] = 0;

for (int j=tx; j<outputSize; j+=i_step)
{
const float grad = blockGradOutput[j];
buffer[tx] += blockOutput[j]*grad;
_blockGradOutput[j] = grad*outputScale;
}

// add (reduce)
for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
{
__syncthreads();
if (tx < stride)
buffer[tx] += buffer[tx+stride];
}

if (tx == 0)
gradOutputScale_k[m] = buffer[0]/(outputScale+0.00000001);
}
}