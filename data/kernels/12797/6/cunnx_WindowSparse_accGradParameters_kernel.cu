#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputWindowSize, int outputWindowSize, int inputSize, int outputSize, float scale)
{
__shared__ float buffer[WINDOWSPARSE_THREADS];
int tx = threadIdx.x;
int i_step = blockDim.x;
int k = blockIdx.x;

int inputIdx = (int)inputIndice[k] - 1;
int outputIdx = (int)outputIndice[k] - 1;

float *input_k = input + k*inputWindowSize;
float *gradOutput_k = gradOutput + k*outputWindowSize;
float *gradWeight_k = gradWeight + outputIdx*inputSize + inputIdx;
float *gradBias_k = gradBias + outputIdx;

// addr weights (scalar-products)
for (int i=tx; i<inputWindowSize; i+=i_step)
{
// copy input to buffer
buffer[tx] = input_k[i]*scale;

// multiply accumulate weights
for (int j=0; j<outputWindowSize; j++)
atomicAdd(&(gradWeight_k[j*inputSize + i]), gradOutput_k[j]*buffer[tx]);
}

// cadd bias i.e. multiply accumulate biases
for (int j=tx; j<outputWindowSize; j+=i_step)
atomicAdd(&(gradBias_k[j]), gradOutput_k[j]*scale);
}