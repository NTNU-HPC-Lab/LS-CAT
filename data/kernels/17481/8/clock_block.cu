#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clock_block(clock_t *d, clock_t clock_count) {
clock_t start_clock = clock64();
clock_t clock_offset = 0;
while (clock_offset < clock_count) {
clock_offset = clock64() - start_clock;
}
if (d) {
*d = clock_offset;
}
}