#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

// Thread block size
#define BLOCK_SIZE 16

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
/* ------------------ Cuda Code --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
* defined in the beginning of this code.  B[][] is initialized to zeros.
*/




/* returns a seed for srand based on the time */
__global__ void matrixNorm(float* d_in, float* d_out, float* d_mean, float* d_sd, int N)
{
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

unsigned int i = idx_y * N + idx_x;

if (d_sd[blockIdx.y] == 0.0)
d_out[i] = 0.0;
else
d_out[i] = (d_in[i] - d_mean[blockIdx.x]) / d_sd[blockIdx.x];
}