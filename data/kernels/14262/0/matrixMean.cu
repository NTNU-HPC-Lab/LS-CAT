#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

// Thread block size
#define BLOCK_SIZE 16

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
/* ------------------ Cuda Code --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
* defined in the beginning of this code.  B[][] is initialized to zeros.
*/




/* returns a seed for srand based on the time */
__global__ void matrixMean(float* d_in, float* d_mean, int N)
{
extern __shared__ float sdata[];

//each thread loads one element from global to shared mem
int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

unsigned int tid = threadIdx.y;
unsigned int i = idx_y * N + idx_x;
sdata[tid] = d_in[i];
__syncthreads();

// do reduction in shared mem
for(unsigned int s=1; s < blockDim.y; s *= 2)
{
if(tid +s < N)
{
if(tid % (2*s) == 0)
{
sdata[tid] += sdata[tid + s];
}
}
__syncthreads();
}

// write result for this block to global mem
if(tid == 0)
{
d_mean[blockIdx.x] = sdata[0]/(float) N;
}
}