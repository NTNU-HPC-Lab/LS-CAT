#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _weightBRightkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float*dev_wg3, float *dev_recon)
{
int ind0, indg[3],q;
int k=blockIdx.x*blockDim.x + threadIdx.x;
if (k>=num_slices)
return;
ind0 = (num_grid-1) + (num_grid-1)*num_grid + k*num_grid*num_grid;

indg[0] = ind0-1;
indg[1] = ind0-num_grid;
indg[2] = ind0-num_grid-1;

for (q = 0; q < 3; q++) {
dev_F[ind0] += 2*beta*dev_wg3[q];
dev_G[ind0] -= 2*beta*dev_wg3[q]*(dev_recon[ind0]+dev_recon[indg[q]]);
}
}