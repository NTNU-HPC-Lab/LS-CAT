#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _weightLeftkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float*dev_wg5, float *dev_recon)
{
uint q;
int ind0, indg[5];
uint k = blockIdx.x*blockDim.x + threadIdx.x;
uint n = blockIdx.y*blockDim.y + threadIdx.y+1;

if ((k>=num_slices)||(n<1)||(n>=(num_grid-1)))
return;

ind0 = n*num_grid + k*num_grid*num_grid;

indg[0] = ind0+1;
indg[1] = ind0+num_grid;
indg[2] = ind0-num_grid;
indg[3] = ind0+num_grid+1;
indg[4] = ind0-num_grid+1;

for (q = 0; q < 5; q++) {
dev_F[ind0] += 2*beta*dev_wg5[q];
dev_G[ind0] -= 2*beta*dev_wg5[q]*(dev_recon[ind0]+dev_recon[indg[q]]);
}
}