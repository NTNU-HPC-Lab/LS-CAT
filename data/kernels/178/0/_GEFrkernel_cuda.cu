#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _GEFrkernel_cuda(int num_slices,int num_grid,float* dev_recon,float* dev_G,float* dev_E,float* dev_F)
{
uint m = blockIdx.x*blockDim.x + threadIdx.x;
uint n = blockIdx.y*blockDim.y + threadIdx.y;
uint k = blockIdx.z;
uint i = m + n*num_grid + k*num_grid*num_grid;
if((m>=num_grid)||(n>=num_grid)||(k>=num_slices))
return;
//	int i = m + n*num_grid + k*num_grid*num_grid;
//  recon[i] = (-G[i]+sqrt(G[i]*G[i]-8*E[i]*F[i]))/(4*F[i]);
dev_recon[i] = (-dev_G[i]+sqrtf(dev_G[i]*dev_G[i]-8.*dev_E[i]*dev_F[i]))/(4.*dev_F[i]);
}