#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _weightInnerkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float*dev_wg8, float *dev_recon)
{
uint m = blockIdx.x*blockDim.x + threadIdx.x+1;
uint n = blockIdx.y*blockDim.y + threadIdx.y+1;
uint k = blockIdx.z;
int q;
int ind0, indg[8];

if ((k>=num_slices)||(n<1)||(n>=(num_grid-1))||(m<1)||(m>=(num_grid-1)))
return;

ind0 = m + n*num_grid + k*num_grid*num_grid;

indg[0] = ind0+1;
indg[1] = ind0-1;
indg[2] = ind0+num_grid;
indg[3] = ind0-num_grid;
indg[4] = ind0+num_grid+1;
indg[5] = ind0+num_grid-1;
indg[6] = ind0-num_grid+1;
indg[7] = ind0-num_grid-1;

for (q = 0; q < 8; q++) {
dev_F[ind0] += 2*beta*dev_wg8[q];
dev_G[ind0] -= 2*beta*dev_wg8[q]*(dev_recon[ind0]+dev_recon[indg[q]]);
}
}