#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "_kernelpp_cuda.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int num_projections = 1;
float mov = 1;
int num_pixels = 1;
int num_grid = 1;
int num_slices = 1;
float *dev_gridx = NULL;
hipMalloc(&dev_gridx, XSIZE*YSIZE);
float *dev_gridy = NULL;
hipMalloc(&dev_gridy, XSIZE*YSIZE);
float *dev_suma = NULL;
hipMalloc(&dev_suma, XSIZE*YSIZE);
float *dev_E = NULL;
hipMalloc(&dev_E, XSIZE*YSIZE);
float *dev_data = NULL;
hipMalloc(&dev_data, XSIZE*YSIZE);
float *dev_recon = NULL;
hipMalloc(&dev_recon, XSIZE*YSIZE);
float *dev_theta = NULL;
hipMalloc(&dev_theta, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
_kernelpp_cuda<<<gridBlock,threadBlock>>>(num_projections,mov,num_pixels,num_grid,num_slices,dev_gridx,dev_gridy,dev_suma,dev_E,dev_data,dev_recon,dev_theta);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
_kernelpp_cuda<<<gridBlock,threadBlock>>>(num_projections,mov,num_pixels,num_grid,num_slices,dev_gridx,dev_gridy,dev_suma,dev_E,dev_data,dev_recon,dev_theta);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
_kernelpp_cuda<<<gridBlock,threadBlock>>>(num_projections,mov,num_pixels,num_grid,num_slices,dev_gridx,dev_gridy,dev_suma,dev_E,dev_data,dev_recon,dev_theta);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}