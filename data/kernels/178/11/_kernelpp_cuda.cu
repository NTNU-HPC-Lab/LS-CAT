#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _kernelpp_cuda(int num_projections, float mov, int num_pixels, int num_grid, int num_slices, float* dev_gridx, float* dev_gridy, float* dev_suma, float * dev_E, float* dev_data, float * dev_recon, float* dev_theta){
uint q = blockIdx.x*blockDim.x + threadIdx.x;
uint m = blockIdx.y*blockDim.y + threadIdx.y;
const double PI = 3.141592653589793238462;
bool quadrant;
float sinq, cosq;
float xi, yi;
float srcx, srcy, detx, dety;
float slope, islope;
int n,i,j,k;
int alen, blen, len;
int i1, i2;
float x1, x2;
int indx, indy;
int io;
float midx, midy, diffx, diffy;
float simdata;
float upd;
float coordx[MAX_NUM_GRID];
float coordy[MAX_NUM_GRID];
float ax[MAX_NUM_GRID];
float ay[MAX_NUM_GRID];
float bx[MAX_NUM_GRID];
float by[MAX_NUM_GRID];
float coorx[MAX_NUM_GRID*2];
float coory[MAX_NUM_GRID*2];
float leng[MAX_NUM_GRID*2];
int indi[MAX_NUM_GRID*2];
if((m>=num_pixels)||(q>=num_projections))
return;


// Calculate the sin and cos values
// of the projection angle and find
// at which quadrant on the cartesian grid.
sinq = sin(dev_theta[q]);
cosq =  cos(dev_theta[q]);
if ((dev_theta[q] >= 0 && dev_theta[q] < PI/2) ||
(dev_theta[q] >= PI && dev_theta[q] < 3*PI/2)) {
quadrant = true;
} else {
quadrant = false;
}

// Find the corresponding source and
// detector locations for a given line
// trajectory of a projection (Projection
// is specified by sinq and cosq).
xi = -1e6;
yi = -(num_pixels-1)/2.+m+mov;
srcx = xi*cosq-yi*sinq;
srcy = xi*sinq+yi*cosq;
detx = -xi*cosq-yi*sinq;
dety = -xi*sinq+yi*cosq;

// Find the intersection points of the
// line connecting the source and the detector
// points with the reconstruction grid. The
// intersection points are then defined as:
// (coordx, gridy) and (gridx, coordy)
slope = (srcy-dety)/(srcx-detx);
islope = 1/slope;

for (n = 0; n <= num_grid; n++) {
coordx[n] = islope*(dev_gridy[n]-srcy)+srcx;
coordy[n] = slope*(dev_gridx[n]-srcx)+srcy;
}

// Merge the (coordx, gridy) and (gridx, coordy)
// on a single array of points (ax, ay) and trim
// the coordinates that are outside the
// reconstruction grid.
alen = 0;
blen = 0;
for (n = 0; n <= num_grid; n++) {
if (coordx[n] > dev_gridx[0]) {
if (coordx[n] < dev_gridx[num_grid]) {
ax[alen] = coordx[n];
ay[alen] = dev_gridy[n];
alen++;
}
}
if (coordy[n] > dev_gridy[0]) {
if (coordy[n] < dev_gridy[num_grid]) {
bx[blen] = dev_gridx[n];
by[blen] = coordy[n];
blen++;
}
}
}
len = alen+blen;

// Sort the array of intersection points (ax, ay).
// The new sorted intersection points are
// stored in (coorx, coory).
i = 0;
j = 0;
k = 0;
if (quadrant) {
while (i < alen && j < blen)
{
if (ax[i] < bx[j]) {
coorx[k] = ax[i];
coory[k] = ay[i];
i++;
k++;
} else {
coorx[k] = bx[j];
coory[k] = by[j];
j++;
k++;
}
}
while (i < alen) {
coorx[k] = ax[i];
coory[k] = ay[i];
i++;
k++;
}
while (j < blen) {
coorx[k] = bx[j];
coory[k] = by[j];
j++;
k++;
}
} else {
while (i < alen && j < blen)
{
if (ax[alen-1-i] < bx[j]) {
coorx[k] = ax[alen-1-i];
coory[k] = ay[alen-1-i];
i++;
k++;
} else {
coorx[k] = bx[j];
coory[k] = by[j];
j++;
k++;
}
}

while (i < alen) {
coorx[k] = ax[alen-1-i];
coory[k] = ay[alen-1-i];
i++;
k++;
}
while (j < blen) {
coorx[k] = bx[j];
coory[k] = by[j];
j++;
k++;
}
}

// Calculate the distances (leng) between the
// intersection points (coorx, coory). Find
// the indices of the pixels on the
// reconstruction grid (indi).
for (n = 0; n < len-1; n++) {
diffx = coorx[n+1]-coorx[n];
diffy = coory[n+1]-coory[n];
leng[n] = sqrt(diffx*diffx+diffy*diffy);
midx = (coorx[n+1]+coorx[n])/2;
midy = (coory[n+1]+coory[n])/2;
x1 = midx+num_grid/2.;
x2 = midy+num_grid/2.;
i1 = (int)(midx+num_grid/2.);
i2 = (int)(midy+num_grid/2.);
indx = i1-(i1>x1);
indy = i2-(i2>x2);
indi[n] = indx+indy*num_grid;
}

// Note: The indices (indi) and the corresponding
// weights (leng) are the same for all slices. So,
// there is no need to calculate them for each slice.


//*******************************************************
// Below is for updating the reconstruction grid.

for (n = 0; n < len-1; n++) {
//        suma[indi[n]] += leng[n];
atomicAdd(&(dev_suma[indi[n]]),leng[n]);
}

for (k = 0; k < num_slices; k++) {
i = k*num_grid*num_grid;
io = m + k*num_pixels + q*num_slices*num_pixels;

simdata = 0;
for (n = 0; n < len-1; n++) {
simdata += dev_recon[indi[n]+i] * leng[n];
}
upd = dev_data[io]/simdata;
for (n = 0; n < len-1; n++) {
//            E[indi[n]+i] -= dev_recon[indi[n]+i]*upd*leng[n];
atomicAdd(&(dev_E[indi[n]+i]),-dev_recon[indi[n]+i]*upd*leng[n]);
}
}
}