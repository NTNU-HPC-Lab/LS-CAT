#include "hip/hip_runtime.h"
#include "includes.h"
/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/



#define blockx 16
#define blocky 16


__global__ void _sGkernel_cuda(int num_slices, int num_grid, float* dev_G,float *dev_suma)
{
uint m = blockIdx.x*blockDim.x + threadIdx.x;
uint n = blockIdx.y*blockDim.y + threadIdx.y;
uint k = blockIdx.z;
uint i = m + n*num_grid + k*num_grid*num_grid;
uint j = m + n*num_grid;
if((m>=num_grid)||(n>=num_grid)||(k>=num_slices))
return;
//	G[k*num_grid*num_grid+n] += suma[n];
dev_G[i]+= dev_suma[j];
}