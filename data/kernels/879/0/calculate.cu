#include "hip/hip_runtime.h"
#include "includes.h"
// %%cu
// as data type is int, sum might overflow (depending on rand(), but the seq and parallel answers are still equal, or change int to long long (too lazy sorry))
#define THREADS_PER_BLOCK 256
using namespace std;

__global__ void calculate(int *arr_in, int* arr_out, int sz, int option){
int ind = threadIdx.x;
int dim = blockDim.x;
extern __shared__ int shared_mem[];
int actual_ind = blockIdx.x*blockDim.x + ind;
if(actual_ind < sz){
shared_mem[ind] = arr_in[actual_ind];
}else{
if(option == 0 || option == 3)
shared_mem[ind] = 0;
else if(option == 1){//maximum
shared_mem[ind] = -INT_MAX;
}else{//minimum
shared_mem[ind] = INT_MAX;
}
}
__syncthreads();
for(int i=dim/2 ; i > 0 ; i=i/2){
if(ind<i){
if(option == 0 || option == 3)
shared_mem[ind]+=shared_mem[ind+i];
else if(option == 1){
shared_mem[ind]=max(shared_mem[ind],shared_mem[ind+i]);
}else{
shared_mem[ind]=min(shared_mem[ind],shared_mem[ind+i]);
}
}
__syncthreads();
}
arr_out[blockIdx.x]=shared_mem[0];
}