#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multVector(int *d1_in, int *d2_in, int *d_out, int n, int m){
int ind = blockDim.x*blockIdx.x + threadIdx.x;
if(ind<m){
d_out[ind]=0;
for(int i=0;i<n;i++){
d_out[ind]+= d1_in[i]*d2_in[i*m+ind];
}
}
}