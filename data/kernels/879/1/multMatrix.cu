#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multMatrix(int *d1_in, int *d2_in, int *d_out, int n, int m, int k){
int indx = threadIdx.x;
int indy = threadIdx.y;
int ind = indy*k+indx;
//printf("%d %d\n",indy,indx);
if(ind<n*k){
d_out[ind] = 0;
for(int i=0;i<m;i++){
d_out[ind] += d1_in[indy*m+i]*d2_in[i*k+indx];
}
}
}