#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVector(int *d1_in, int *d2_in, int *d_out, int n){
int ind = blockDim.x*blockIdx.x + threadIdx.x;
if(ind<n){
d_out[ind] = d1_in[ind]+d2_in[ind];
}
}