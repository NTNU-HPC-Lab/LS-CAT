#include "hip/hip_runtime.h"
#include "includes.h"


#define ARRAY_SIZE 200
#define ARRAY_BYTES ARRAY_SIZE * sizeof(float)


__global__ void CalculateSquare(float* p_out, float* p_in)
{
int index = threadIdx.x;
float valueToSuqare = p_in[index];
p_out[index] = valueToSuqare * valueToSuqare;
}