#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SoftmaxKernel( float *outputPtr, float expSum, int layerSize )
{
// i: neuron id
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < layerSize)
{
// exp value is already present in the output array, so just divide by sum of exps (computed before kernel call)
outputPtr[i] /= expSum;
}


}