#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float Dist_between_two_vec(float * v0, float *v1, int size) {
float dist = 0;
for (int i = 0; i < size; i++)
dist += (v0[i] - v1[i])*(v0[i] - v1[i]);

return sqrt(dist);
}
__global__ void Dist_between_two_vec_naive(float * v0, float *v1, int size, float * dst) {
float dist = 0;
for (int i = 0; i < size; i++)
dist += (v0[i] - v1[i]);//*(v0[i]-v1[i]);

dst[0] = dist;
}