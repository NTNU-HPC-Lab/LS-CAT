#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InvolveVector(float* input, float* output, int inputSize)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize - 1)
{
output[0] = input[0];
output[threadId + 1] = input[inputSize - threadId - 1];
}
}