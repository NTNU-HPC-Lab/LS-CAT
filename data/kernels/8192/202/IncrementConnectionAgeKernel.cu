#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IncrementConnectionAgeKernel(  int cell, int *connection, int *age, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
if(connection[cell * maxCells + threadId] == 1)
{
age[cell * maxCells + threadId] += 1;
age[threadId * maxCells + cell] += 1;
}

}
}