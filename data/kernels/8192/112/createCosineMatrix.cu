#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createCosineMatrix(float* matrix, int xsize){
int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
+ threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

int i;
for (i = 0; i < xsize; i++){
if (threadGlobalID == 0)
matrix[threadGlobalID + i * xsize] = 1 / sqrt((float)xsize);
else
matrix[threadGlobalID + i * xsize] = (sqrt((float)2 / xsize) * cos((PI * (2 * i + 1) * threadGlobalID) / (2 * xsize)));
}
}