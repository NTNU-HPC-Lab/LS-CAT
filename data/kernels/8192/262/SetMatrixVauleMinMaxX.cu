#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetMatrixVauleMinMaxX( float* matrix, int cols, int size, int id_min, int id_max, float value)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
int id_column = id%cols;
if (id_column >= id_min && id_column <= id_max && id < size)
matrix[id] = value;
}