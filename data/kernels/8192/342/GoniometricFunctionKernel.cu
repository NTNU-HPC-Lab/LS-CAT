#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GoniometricFunctionKernel(float* input, float* output, const int size, const int type)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
if(id < size)
{	 // Sine = 0, Cosine = 1, Tan = 2, Tanh = 3, Sinh = 4, Cosh = 5  see MyGonioType in MyTransform.cs
switch (type)
{
case 0:
output[id] = sinf(input[id]);
break;
case 1:
output[id] = cosf(input[id]);
break;
case 2:
output[id] = tanf(input[id]);
break;
case 3:
output[id] = tanhf(input[id]);
break;
case 4:
output[id] = sinhf(input[id]);
break;
case 5:
output[id] = coshf(input[id]);
break;
case 6:
output[id] = asinf(input[id]);
break;
case 7:
output[id] = acosf(input[id]);
break;
case 10:
output[id] = atan2f(input[2*id], input[2*id+1]);
break;
}
}
}