#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TwoNodesDistanceKernel(  float *twoNodesDifference, float *twoNodesDistance, int vectorLength  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
float sum = 0.00f;
float value;
for(int i = 0; i < vectorLength; i++)
{
value = twoNodesDifference[threadId * vectorLength + i];
sum += value*value;
}
twoNodesDistance[threadId] = sqrtf(sum);
}
}