#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grow(float *matrices, const int dimension, const int coefficients, const int population, float *chromosomes, const float * noise, const float mutationRate, const int kept, const float* fitnesses, int *mark, const int alpha){

int i, wloc;

hiprandState st;

// For up to a 1D grid of 3D blocks...
int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
+ threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

int chromOffset = threadGlobalID * coefficients;
int parent1, parent2, point;
float tmp1, tmp2;

// Init the random number generator
hiprand_init((int)noise[threadGlobalID] << threadGlobalID, threadGlobalID * (threadGlobalID == population - 1 ? noise[0] : noise[threadGlobalID]), 0, &st);

// Repopulate
// The threads with the keepmask are kept, all others are replaced with crossovers
if (threadGlobalID > kept - 1){
// pick two parents -- 0 is not included in the random distribution
parent1 = floor(hiprand_uniform(&st) * kept);
parent2 = floor(hiprand_uniform(&st) * kept);
//pick a point on the chromosome
point = floor(hiprand_uniform(&st) * coefficients);
for (i = 0; i < point; i++){
chromosomes[chromOffset + i] = chromosomes[parent1 * coefficients + i];
}
//Copy past the point for parent 2
for (i = point; i < coefficients; i++){
chromosomes[chromOffset + i] = chromosomes[parent2 * coefficients + i];
}
}

// Mutate children
if (threadGlobalID > kept - 1){
for (i = 0; i < coefficients; i++){
if (hiprand_uniform(&st) <= mutationRate){
if (hiprand_uniform(&st) < 0.5){
chromosomes[chromOffset + i] = hiprand_uniform(&st) * -1 * alpha;
}
else{
chromosomes[chromOffset + i] = hiprand_uniform(&st) * alpha;
}
}
}
}

// Permute
if (threadGlobalID < coefficients){
// Mark genes for permutation
for (i = 0; i < population; i++){
if (hiprand_uniform(&st) < (1 - sqrt((fitnesses[i] - fitnesses[population - 1]) / (fitnesses[0] - fitnesses[population - 1])))){
mark[coefficients * i + threadGlobalID] = 1;
}
else{
mark[coefficients * i + threadGlobalID] = 0;
}
}

wloc = -1;
// Permute selected genes
for (i = 0; i < population; i++){
if (mark[coefficients * i + threadGlobalID] == 1){
if (wloc == -1){
wloc = i;
tmp1 = chromosomes[coefficients * i + threadGlobalID];
}
else{
tmp2 = chromosomes[coefficients * i + threadGlobalID];
chromosomes[coefficients * i + threadGlobalID] = tmp1;
tmp1 = tmp2;
}
}
}
if (wloc != -1){
chromosomes[coefficients * wloc + threadGlobalID] = tmp1;
}
}

__syncthreads();
//Place into relevant matrix
for (i = 0; i < dimension*dimension; i++){
matrices[threadGlobalID * dimension * dimension + i] = 0.0f;
}
}