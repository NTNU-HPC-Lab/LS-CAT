#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeDistanceKernel( float *symbolVectors, float *inputVector, float *distance, int symbolSize, int symbols )
{
int symbolId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(symbolId < symbols)
{
float sum = 0.00f;
for(int i = 0; i < symbolSize; i++)
{
sum += symbolVectors[symbolId * symbolSize + i] * inputVector[i];
}
distance[symbolId] = sum;
}
}