#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyConnectionsCoordinatesKernel(   int *connectionMatrix, float *pointsCoordinates, float *vertexData, int *connectionCount, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells * maxCells)
{
if(connectionMatrix[threadId] == 1)
{
int from = threadId / maxCells;
int to = threadId % maxCells;

if(to > from)
{
//int vertexDataOffset = maxCells * 3;
int vertexDataOffset = 0;
int connIdx = atomicAdd( &connectionCount[0], 1);

vertexData[vertexDataOffset + connIdx * 6] = pointsCoordinates[from * 3];
vertexData[vertexDataOffset + connIdx * 6 + 1] = pointsCoordinates[from * 3 + 1];
vertexData[vertexDataOffset + connIdx * 6 + 2] = pointsCoordinates[from * 3 + 2];

vertexData[vertexDataOffset + connIdx * 6 + 3] = pointsCoordinates[to * 3];
vertexData[vertexDataOffset + connIdx * 6 + 4] = pointsCoordinates[to * 3 + 1];
vertexData[vertexDataOffset + connIdx * 6 + 5] = pointsCoordinates[to * 3 + 2];
}

}

}
}