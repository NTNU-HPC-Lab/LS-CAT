#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InterpolateSymbolsKernel( float *symbolVectors, int symbolOneId, int symbolTwoId, float weightOne, float weightTwo, float *resultSymbol, int symbolSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < symbolSize)
{
int symbolOneCellId = symbolOneId * symbolSize + threadId;
int symbolTwoCellId = symbolTwoId * symbolSize + threadId;

resultSymbol[threadId] = weightOne * symbolVectors[symbolOneCellId] + weightTwo * symbolVectors[symbolTwoCellId];
}

}