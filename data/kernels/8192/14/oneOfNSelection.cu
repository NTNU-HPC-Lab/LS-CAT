#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void oneOfNSelection(float *buffer, int* index, int size, float value)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x						//blocks preceeding current block
+ threadIdx.x;

if (threadId < size && threadId != index[0])
{
buffer[threadId] = 0;

}
else if (threadId < size && threadId == index[0]){
buffer[threadId] = value;
}
}