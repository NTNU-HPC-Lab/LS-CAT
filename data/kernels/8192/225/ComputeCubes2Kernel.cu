#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeCubes2Kernel(  float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, float *cubeOperation, float *cubeTexCoordinates, int *activityFlag, float textureWidth, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells * 6)
{
int cellId = threadId / 6;
float fCellId = (float)cellId;
int sideId = threadId % 6;

float x = pointsCoordinates[cellId * 3];
float y = pointsCoordinates[cellId * 3 + 1];
float z = pointsCoordinates[cellId * 3 + 2];

float halfSide = (activityFlag[cellId] == 1) * 0.50f * cubeSide;

int textureOffset = quadOffset + maxCells * 4 * 6 * 3;
float textureAbsLength = (float)maxCells * textureWidth;


vertexData[quadOffset + cellId * 72 + 12*sideId]     = x + operationMaskConstant[12*sideId] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 1] = y + operationMaskConstant[12*sideId + 1] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 2] = z + operationMaskConstant[12*sideId + 2] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 3] = x + operationMaskConstant[12*sideId + 3] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 4] = y + operationMaskConstant[12*sideId + 4] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 5] = z + operationMaskConstant[12*sideId + 5] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 6] = x + operationMaskConstant[12*sideId + 6] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 7] = y + operationMaskConstant[12*sideId + 7] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 8] = z + operationMaskConstant[12*sideId + 8] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 9]  = x + operationMaskConstant[12*sideId + 9] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 10] = y + operationMaskConstant[12*sideId + 10] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 11] = z + operationMaskConstant[12*sideId + 11] * halfSide;


vertexData[textureOffset + cellId * 48 + 8 * sideId]     = ((fCellId + cubeTexCoordinatesConstant[sideId * 8])* textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 1] = cubeTexCoordinatesConstant[sideId * 8 + 1];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 2] = ((fCellId + cubeTexCoordinatesConstant[sideId * 8 + 2]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 3] = cubeTexCoordinatesConstant[sideId * 8 + 3];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 4] = ((fCellId + cubeTexCoordinatesConstant[sideId * 8 + 4]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 5] = cubeTexCoordinatesConstant[sideId * 8 + 5];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 6] = ((fCellId + cubeTexCoordinatesConstant[sideId * 8 + 6]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 7] = cubeTexCoordinatesConstant[sideId * 8 + 7];


/*
vertexData[quadOffset + cellId * 72 + 12*sideId]     = x + cubeOperation[12*sideId] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 1] = y + cubeOperation[12*sideId + 1] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 2] = z + cubeOperation[12*sideId + 2] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 3] = x + cubeOperation[12*sideId + 3] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 4] = y + cubeOperation[12*sideId + 4] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 5] = z + cubeOperation[12*sideId + 5] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 6] = x + cubeOperation[12*sideId + 6] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 7] = y + cubeOperation[12*sideId + 7] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 8] = z + cubeOperation[12*sideId + 8] * halfSide;

vertexData[quadOffset + cellId * 72 + 12*sideId + 9]  = x + cubeOperation[12*sideId + 9] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 10] = y + cubeOperation[12*sideId + 10] * halfSide;
vertexData[quadOffset + cellId * 72 + 12*sideId + 11] = z + cubeOperation[12*sideId + 11] * halfSide;


vertexData[textureOffset + cellId * 48 + 8 * sideId]     = ((fCellId + cubeTexCoordinates[sideId * 8])* textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 1] = cubeTexCoordinates[sideId * 8 + 1];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 2] = ((fCellId + cubeTexCoordinates[sideId * 8 + 2]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 3] = cubeTexCoordinates[sideId * 8 + 3];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 4] = ((fCellId + cubeTexCoordinates[sideId * 8 + 4]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 5] = cubeTexCoordinates[sideId * 8 + 5];

vertexData[textureOffset + cellId * 48 + 8 * sideId + 6] = ((fCellId + cubeTexCoordinates[sideId * 8 + 6]) * textureWidth) / textureAbsLength;
vertexData[textureOffset + cellId * 48 + 8 * sideId + 7] = cubeTexCoordinates[sideId * 8 + 7];
*/
}
}