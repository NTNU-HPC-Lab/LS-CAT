#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMDeltaKernelBPTT( float* deltas, float* cellStates, float* previousCellStates, float* cellStateErrors, float* nextCellStateErrors,  float* outputGateDeltas, float* forgetGateDeltas, float* nextForgetGateDeltas, float* inputGateDeltas, float* nextInputGateDeltas, float* cellInputDeltas,  float* cellInputActivations, float* cellStateActivations, float* outputGateActivations, float* nextForgetGateActivations, float* inputGateActivations,  float* cellInputActivationDerivatives, float* cellStateActivationDerivatives, float* outputGateActivationDerivatives, float* forgetGateActivationDerivatives, float* inputGateActivationDerivatives,  float* cellInputWeights, float* outputGateWeights, float* forgetGateWeights, float* inputGateWeights,  int inputCount, int cellCount, int cellsPerBlock )
{
int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (memoryBlockId < cellCount / cellsPerBlock)
{
outputGateDeltas[memoryBlockId] = 0;
for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
{
outputGateDeltas[memoryBlockId] += cellStateActivations[cellId] * deltas[cellId];
}
outputGateDeltas[memoryBlockId] *= outputGateActivationDerivatives[memoryBlockId];

for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
{
int relativeCellId = cellId - (memoryBlockId * cellsPerBlock);
int peepHoleWeightId = (memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount + relativeCellId;
cellStateErrors[cellId] = deltas[cellId] * outputGateActivations[memoryBlockId] * cellStateActivationDerivatives[cellId] +
nextCellStateErrors[cellId] * nextForgetGateActivations[memoryBlockId] +
nextInputGateDeltas[memoryBlockId] * inputGateWeights[peepHoleWeightId] +
nextForgetGateDeltas[memoryBlockId] * forgetGateWeights[peepHoleWeightId] +
outputGateDeltas[memoryBlockId] * outputGateWeights[peepHoleWeightId];

cellInputDeltas[cellId] = inputGateActivations[memoryBlockId] * cellInputActivationDerivatives[cellId] * cellStateErrors[cellId];
}

inputGateDeltas[memoryBlockId] = 0;
forgetGateDeltas[memoryBlockId] = 0;
for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
{
inputGateDeltas[memoryBlockId] += cellStateErrors[cellId] * cellInputActivations[cellId];
forgetGateDeltas[memoryBlockId] += cellStateErrors[cellId] * previousCellStates[cellId];
}
inputGateDeltas[memoryBlockId] *= inputGateActivationDerivatives[memoryBlockId];
forgetGateDeltas[memoryBlockId] *= forgetGateActivationDerivatives[memoryBlockId];
}
}