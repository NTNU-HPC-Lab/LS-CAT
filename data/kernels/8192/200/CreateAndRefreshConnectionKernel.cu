#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CreateAndRefreshConnectionKernel(  int s1, int s2, int *connection, int *age, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
connection[s1 * maxCells + s2] = 1;
age[s1 * maxCells + s2] = 0;
connection[s2 * maxCells + s1] = 1;
age[s2 * maxCells + s1] = 0;
}
}