#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianSamplePrior(float* input, int inputCount, float* mins, float* maxes, float* randomUniform)
{
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < inputCount)
{
float diff = maxes[i] - mins[i];
input[i] = randomUniform[i] * diff + mins[i];
}
}