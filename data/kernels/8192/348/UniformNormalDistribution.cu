#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UniformNormalDistribution(float *from, float *to, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

float tmp;

if (id < size)
{
tmp = normcdf(from[id] * sqrt((float)size));

to[id] = (tmp -0.5)*2;
}
}