#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedUpdateParametersKernel( float *weightPtr, float *biasPtr, float *weightLearningRatePtr, float *biasLearningRatePtr, float *weightGradientPtr, float *biasGradientPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
if (!dropoutMaskPtr[j])
{
// update weights
int index = j;
for (i = 0; i < prevLayerSize; i++)
{
weightPtr[index] -= weightLearningRatePtr[index] * weightGradientPtr[index];

index += thisLayerSize;
}

// update bias
biasPtr[j] -= biasLearningRatePtr[j] * biasGradientPtr[j];
}
}
}