#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Sum(float * A, float  *B, float *C, int size) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size) {
C[id] = A[id] + B[id];
}
}