#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeOutputWeightsKernel( float *outputWeights, float *outputWeightDeltas, float *outputDeltas, float *hiddenActivations,  float trainingRate, float momentum )
{
int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

int to = weightId / D_HIDDEN_UNITS;
int from = weightId % D_HIDDEN_UNITS;

if (weightId < D_OUTPUT_UNITS * D_HIDDEN_UNITS)
{
float gradient = outputDeltas[to] * hiddenActivations[from];
float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
outputWeightDeltas[weightId] = weightDelta;
outputWeights[weightId] += weightDelta;
}
}