#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeEuklidianDistancesKernel(  float *inputImg, int imgWidth, int imgHeight, float *centroidCoordinates, float *distanceMatrix, int centroids, int inputSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
int pointX = threadId % imgWidth;
int pointY = threadId / imgWidth;

float X = (float)pointX;
float Y = (float)pointY;

float dist;

float centroidX;
float centroidY;

for(int c = 0; c < centroids; c++)
{
centroidX = centroidCoordinates[c * 2];
centroidY = centroidCoordinates[c * 2 + 1];

dist = sqrtf( (centroidX - X) * (centroidX - X) + (centroidY - Y) * (centroidY - Y) );
distanceMatrix[c * inputSize + threadId] = dist;
}
}
}