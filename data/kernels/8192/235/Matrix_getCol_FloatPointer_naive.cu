#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_getCol_FloatPointer_naive(const float * A , int Acount, int Acols, const float * colId , int empty_par1, int empty_par2, float * out0 , int out0count, int out0cols)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<Acount/Acols)
{
out0[id] = A[id*Acols + (int)colId[0]];
}
}