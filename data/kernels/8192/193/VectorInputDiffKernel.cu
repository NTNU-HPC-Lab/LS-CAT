#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorInputDiffKernel(  float *input, int inputSize, float *referenceVector, int maxCells, float *difference  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells * inputSize)
{
difference[threadId] = input[threadId % inputSize] - referenceVector[threadId];
}
}