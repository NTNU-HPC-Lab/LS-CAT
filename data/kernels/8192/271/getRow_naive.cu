#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getRow_naive(const float * A, float * row_id, float * out, int Acols) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < Acols) {
out[id] = A[id + (int)(*row_id)*Acols];
}
}