#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_getRow_FloatPointer_naive(const float * A , int Acount, int Acols, const float * rowId , int empty_par1, int empty_par2, float * out0 , int out0count, int out0cols)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<Acols)
{
out0[id] = A[id + (int)rowId[0]*Acols];
}
}