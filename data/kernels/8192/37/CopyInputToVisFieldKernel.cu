#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyInputToVisFieldKernel(  float *input, float *visField,  int inputSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
visField[threadId] = input[threadId];
}
}