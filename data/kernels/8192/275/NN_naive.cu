#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float Dist_between_two_vec(float * v0, float *v1, int size) {
float dist = 0;
for (int i = 0; i < size; i++)
dist += (v0[i] - v1[i])*(v0[i] - v1[i]);

return sqrt(dist);
}
__global__ void NN_naive(float * A, int colsA, int sizeA, float * B, int colsB, int numsB, int dim, float * idx, float * dist) {
float tmp_dist = 99999;
int nn_id = -1;
int idA = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;

for (int idB = 0; idB < (numsB*colsB); idB += colsB) {
float adist = Dist_between_two_vec(A + colsA*idA, B + colsB*idB, dim);
if (tmp_dist > adist) {
tmp_dist = adist;
nn_id = idB;
}
}
*(dist + idA) = tmp_dist;
*(idx + idA) = nn_id;
}