#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedAdjustMemoryKernel( float *weightsGradPtr, float *biasGradPtr, float *weightGradCurvePtr, float *biasGradCurvePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
if (!dropoutMaskPtr[j])
{
int index = j;
for (i = 0; i < prevLayerSize; i++)
{
// check for weight outliers
if (
abs(weightsGradPtr[index] - avgWeightGradPtr[index]) > 2 * sqrtf(avgWeightGradVarPtr[index]) - avgWeightGradPtr[index] * avgWeightGradPtr[index] ||
abs(weightGradCurvePtr[index] - avgWeightGradCurvePtr[index]) > 2 * sqrtf(avgWeightGradCurveVarPtr[index] - avgWeightGradCurveVarPtr[index] * avgWeightGradCurveVarPtr[index])
)
// TODO: test which one works best
//weightMemorySizePtr[index] += 1; // original method suggested in http://arxiv.org/pdf/1301.3764.pdf
weightMemorySizePtr[index] = 2.2f; // reset to 2.2 according to the Adasecant method in http://arxiv.org/pdf/1412.7419v4.pdf

index += thisLayerSize;
}

// check for bias outliers
if (
abs(biasGradPtr[j] - avgBiasGradPtr[j]) > 2 * sqrtf(avgBiasGradVarPtr[j]) - avgBiasGradPtr[j] * avgBiasGradPtr[j] ||
abs(biasGradCurvePtr[j] - avgBiasGradCurvePtr[j]) > 2 * sqrtf(avgBiasGradCurveVarPtr[j] - avgBiasGradCurveVarPtr[j] * avgBiasGradCurveVarPtr[j])
)
// TODO: test which one works best
//biasMemorySizePtr[j] += 1; // original method suggested in http://arxiv.org/pdf/1301.3764.pdf
biasMemorySizePtr[j] = 2.2f; // reset to 2.2 according to the Adasecant method in http://arxiv.org/pdf/1412.7419v4.pdf
}
}
}