#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CropKernel(float min, float max, float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if(id < size)
{
output[id] = fmaxf(fminf(input[id], max), min);
}
}