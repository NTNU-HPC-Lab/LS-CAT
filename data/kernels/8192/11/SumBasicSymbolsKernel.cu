#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumBasicSymbolsKernel( float *symbolVectors, int symbolOneId, int symbolTwoId, float *result, int symbolSize )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < symbolSize)
{
result[threadId] = symbolVectors[symbolOneId * symbolSize + threadId] + symbolVectors[symbolTwoId * symbolSize + threadId];
}
}