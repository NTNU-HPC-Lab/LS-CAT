#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearValuesKernel(const float min, const float max, float* output, const int size, const int shift)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

__shared__ float delta;

if (threadIdx.x == 0)
delta = (max-min)/fmaxf((size-1), 1);
__syncthreads();

if(id < size)
{
output[(id + shift) % size] = min + id * delta;
}
}