#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, int areaWidth, int areaHeight, float r, float g, float b)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int targetPixels = targetWidth * targetHeight;

int texturePixels = areaWidth * areaHeight;

int idTextureRgb = id / texturePixels;
int idTexturePixel = (id - idTextureRgb * texturePixels); // same as (id % texturePixels), but the kernel runs 10% faster
int idTextureY = idTexturePixel / areaWidth;
int idTextureX = (idTexturePixel - idTextureY * areaWidth); // same as (id % textureWidth), but the kernel runs another 10% faster


if (idTextureRgb < 3) // 3 channels that we will write to
{
// if the texture pixel offset by inputX, inputY, lies inside the target
if (idTextureX + inputX < targetWidth &&
idTextureX + inputX >= 0 &&
idTextureY + inputY < targetHeight &&
idTextureY + inputY >= 0)
{
float color = 0.0f;
switch (idTextureRgb)
{
case 0:
color = r;
break;
case 1:
color = g;
break;
case 2:
color = b;
break;
}
int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
target[tIndex] = color;
}
}
}