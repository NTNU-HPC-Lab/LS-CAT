#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MaskByNaN( float* inputImage, float* mask, float* outputImage, int count ) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;

if (id < count)
{
if (mask[id] == 0.0f)
{
outputImage[id] = NAN;
}
else {
outputImage[id] = inputImage[id];
}
}
}