#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeInputWeightsKernel( float *inputWeights, float *inputWeightDeltas, float *outputWeights, float *outputDeltas, float *inputWeightRTRLDerivatives,  float trainingRate, float momentum )
{
int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS)
{
float gradient = 0;

for (int i = 0; i < D_OUTPUT_UNITS; i++)
{
float sum = 0;
for (int j = 0; j < D_HIDDEN_UNITS; j++)
{
sum += outputWeights[i * D_HIDDEN_UNITS + j] * inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
}

gradient += outputDeltas[i] * sum;
}

float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
inputWeightDeltas[weightId] = weightDelta;
inputWeights[weightId] += weightDelta;
}
}