#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FillAdjacencyMatrix(float* adj_mat , float* maskBuffer , int size , int cols , int rows ,int Nsegs){
int idx = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int icol = idx % cols;
int irow = idx / cols;
int seg_id1=-1;
if (idx<size){
if (icol<cols-2 && irow<rows-2 && irow>1 && icol>1){
seg_id1 = maskBuffer[idx];
if (seg_id1!=maskBuffer[idx+1]){
adj_mat[ (int)maskBuffer[idx+1] + seg_id1*Nsegs ]=1;
adj_mat[ seg_id1 + Nsegs*(int)maskBuffer[idx+1] ]=1; /// it can happen that a->b, but b->a wont appear...
}
else if (seg_id1!=maskBuffer[idx-cols]){
adj_mat[ (int)maskBuffer[idx-cols] + seg_id1*Nsegs ]=1;
adj_mat[ seg_id1 + Nsegs*(int)maskBuffer[idx-cols] ]=1; /// it can happen that a->b, but b->a wont appear...
}
}
}
}