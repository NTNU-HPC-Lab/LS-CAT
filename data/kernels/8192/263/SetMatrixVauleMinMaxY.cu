#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetMatrixVauleMinMaxY( float* matrix, int cols, int size, int id_min, int id_max, float value)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
int id_row = id / cols;
if (id_row >= id_min && id_row <= id_max && id < size)
matrix[id] = value;
}