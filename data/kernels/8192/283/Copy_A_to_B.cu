#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void Copy_A_to_B (float * A , float * B , int size){
int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
if (id<size)
B[id] = A[id];
}