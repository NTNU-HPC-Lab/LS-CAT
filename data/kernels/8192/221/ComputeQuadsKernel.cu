#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeQuadsKernel(  float *pointsCoordinates, float *vertexData, int quadOffset, float textureSide, int *activityFlag, int textureWidth, int maxCells   )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
float x = pointsCoordinates[threadId * 3];
float y = pointsCoordinates[threadId * 3 + 1];
float z = pointsCoordinates[threadId * 3 + 2];

float halfSide = 0.50f * textureSide;
if(activityFlag[threadId] == 0)
{
halfSide = 0.00f;
}

int textureOffset = quadOffset + maxCells * 4 * 3 * 3;
float textureAbsLength = (float)(maxCells * textureWidth);

// vertical x-alligned
vertexData[quadOffset + threadId * 36] = x - halfSide;
vertexData[quadOffset + threadId * 36 + 1] = y + halfSide;
vertexData[quadOffset + threadId * 36 + 2] = z;

vertexData[textureOffset + threadId * 24] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 1] = 0.00f;

vertexData[quadOffset + threadId * 36 + 3] = x - halfSide;
vertexData[quadOffset + threadId * 36 + 4] = y - halfSide;
vertexData[quadOffset + threadId * 36 + 5] = z;

vertexData[textureOffset + threadId * 24 + 2] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 3] = 1.00f;

vertexData[quadOffset + threadId * 36 + 6] = x + halfSide;
vertexData[quadOffset + threadId * 36 + 7] = y - halfSide;
vertexData[quadOffset + threadId * 36 + 8] = z;

vertexData[textureOffset + threadId * 24 + 4] = (float)((threadId+1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 5] = 1.00f;


vertexData[quadOffset + threadId * 36 + 9] = x + halfSide;
vertexData[quadOffset + threadId * 36 + 10] = y + halfSide;
vertexData[quadOffset + threadId * 36 + 11] = z;

vertexData[textureOffset + threadId * 24 + 6] = (float)((threadId+1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 7] = 0.00f;

// horizontal
vertexData[quadOffset + threadId * 36 + 12] = x - halfSide;
vertexData[quadOffset + threadId * 36 + 13] = y;
vertexData[quadOffset + threadId * 36 + 14] = z + halfSide;

vertexData[textureOffset + threadId * 24 + 8] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 9] = 1.00f;

vertexData[quadOffset + threadId * 36 + 15] = x - halfSide;
vertexData[quadOffset + threadId * 36 + 16] = y;
vertexData[quadOffset + threadId * 36 + 17] = z - halfSide;

vertexData[textureOffset + threadId * 24 + 10] = (float)(threadId * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 11] = 0.00f;

vertexData[quadOffset + threadId * 36 + 18] = x + halfSide;
vertexData[quadOffset + threadId * 36 + 19] = y;
vertexData[quadOffset + threadId * 36 + 20] = z - halfSide;

vertexData[textureOffset + threadId * 24 + 12] = (float)((threadId+1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 13] = 0.00f;

vertexData[quadOffset + threadId * 36 + 21] = x + halfSide;
vertexData[quadOffset + threadId * 36 + 22] = y;
vertexData[quadOffset + threadId * 36 + 23] = z + halfSide;

vertexData[textureOffset + threadId * 24 + 14] = (float)((threadId + 1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 15] = 1.00f;

// vertical z-alligned
vertexData[quadOffset + threadId * 36 + 24] = x;
vertexData[quadOffset + threadId * 36 + 25] = y - halfSide;
vertexData[quadOffset + threadId * 36 + 26] = z + halfSide;

vertexData[textureOffset + threadId * 24 + 16] = (float)((threadId+1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 17] = 1.00f;

vertexData[quadOffset + threadId * 36 + 27] = x;
vertexData[quadOffset + threadId * 36 + 28] = y - halfSide;
vertexData[quadOffset + threadId * 36 + 29] = z - halfSide;

vertexData[textureOffset + threadId * 24 + 18] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 19] = 1.00f;

vertexData[quadOffset + threadId * 36 + 30] = x;
vertexData[quadOffset + threadId * 36 + 31] = y + halfSide;
vertexData[quadOffset + threadId * 36 + 32] = z - halfSide;

vertexData[textureOffset + threadId * 24 + 20] = (float)((threadId) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 21] = 0.00f;

vertexData[quadOffset + threadId * 36 + 33] = x;
vertexData[quadOffset + threadId * 36 + 34] = y + halfSide;
vertexData[quadOffset + threadId * 36 + 35] = z + halfSide;

vertexData[textureOffset + threadId * 24 + 22] = (float)((threadId+1) * textureWidth) / textureAbsLength;
vertexData[textureOffset + threadId * 24 + 23] = 0.00f;
}
}