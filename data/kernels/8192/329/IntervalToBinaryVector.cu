#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IntervalToBinaryVector(float input, float* outputs, int steps)
{
int id = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if(id < steps)
{
float fraction = 1.0f / steps;
outputs[id] = input >= fraction * id && input <= fraction * (id + 1);
}
}