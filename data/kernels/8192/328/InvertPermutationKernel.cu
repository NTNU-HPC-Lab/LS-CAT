#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InvertPermutationKernel(float* input, float* output, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if (id >= size)
return;


int temp = __float2int_rn(input[id]);

if (input == output)
__syncthreads();

output[temp] = id;
}