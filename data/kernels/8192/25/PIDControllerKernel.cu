#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PIDControllerKernel(float* input, float* goal, float* output, float* previousError, float* integral)
{
int id = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (id < D_COUNT)
{
float error = input[id] - goal[id];
integral[id] = D_INTEGRAL_DECAY * integral[id] + error;
float derivative = error - previousError[id];

previousError[id] = error;


float out = D_OFFSET + D_PROPORTIONAL_GAIN * error + D_INTEGRAL_GAIN * integral[id] + D_DERIVATIVE_GAIN * derivative;
if (out > D_MAX_OUTPUT)
out = D_MAX_OUTPUT;
if (out < D_MIN_OUTPUT)
out = D_MIN_OUTPUT;

output[id] = out;
}
}