#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PlotObserverScaleDownScaleKernel(float* history, int nbCurves, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if (id >= size)
return;

int baseAddress = 2 * id;
float val1 = history[baseAddress];
float val2 = history[baseAddress + nbCurves];
float average = (val1 + val2) / 2;
history[id] = average;
}