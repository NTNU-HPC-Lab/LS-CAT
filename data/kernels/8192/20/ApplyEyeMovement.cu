#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyEyeMovement(float* currentEye, float* stats, float moveFactor, float scaleFactor, float scaleBase)
{
float sumWeights = stats[4];

if (sumWeights > 0)
{
currentEye[0] = fmaxf(fminf(moveFactor * stats[0], 1), -1);
currentEye[1] = fmaxf(fminf(moveFactor * stats[1], 1), -1);

float variance = sqrtf((stats[2] + stats[3]) * 0.5);

currentEye[2] = fmaxf(fminf(variance * scaleFactor + scaleBase, 1), 0);
}
else
{
currentEye[0] = 0;
currentEye[1] = 0;
currentEye[2] = 1;
}
}