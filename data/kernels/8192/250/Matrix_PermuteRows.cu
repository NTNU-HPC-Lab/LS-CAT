#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_PermuteRows(const float * A , int Acount, int Acols, const float * B , int Bcount, int Bcols, float * out0 , int out0count, int out0cols)
{
int id = blockDim.x*blockIdx.y*gridDim.x	+   blockDim.x*blockIdx.x	  +   threadIdx.x;
int id_row, id_col, id_rowNew;
if (id<Acount)
{
id_row = id/Acols;
id_col = id%Acols;
id_rowNew = B[id_row]*Acols;
out0[id] = A[id_col + id_rowNew];
}
}