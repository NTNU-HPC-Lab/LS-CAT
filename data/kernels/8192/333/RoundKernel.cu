#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RoundKernel(float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x	+ blockDim.x * blockIdx.x	+ threadIdx.x;
if(id < size)
{
output[id] = round(input[id]);
}
}