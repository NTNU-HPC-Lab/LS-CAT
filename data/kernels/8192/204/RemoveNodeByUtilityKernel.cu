#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RemoveNodeByUtilityKernel(  int *connectionMatrix, int *connectionAge, int *activityFlag, float *utility, float utilityConstant, float *localError, int *neuronAge, float *winningFraction, int *winningCount, float maxError, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
if(activityFlag[threadId] == 1)
{
if(utility[threadId] > 0.00f)
{
if( maxError / utility[threadId] > utilityConstant )
{
activityFlag[threadId] = 0;
localError[threadId] = 0.00f;
neuronAge[threadId] = 0;
winningFraction[threadId] = 0.00f;
winningCount[threadId] = 0;
utility[threadId] = 0.00f;

for(int n = 0; n < maxCells; n++)
{
connectionMatrix[threadId * maxCells + n] = 0;
connectionAge[threadId * maxCells + n] = 0;
connectionMatrix[n * maxCells + threadId] = 0;
connectionAge[n * maxCells + threadId] = 0;
}
}
}
}
}
}