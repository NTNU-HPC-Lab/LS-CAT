#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void MatrixCopy(float* in, float *out, int size)
{
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size)
out[id] = in[id];
}