#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyPointsCoordinatesKernel(  float *pointsCoordinates, int *activityFlag, float xNonValid, float yNonValid, float zNonValid, float *dataVertex, int dataVertexOffset, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
float xToCopy = pointsCoordinates[threadId * 3];
float yToCopy = pointsCoordinates[threadId * 3 + 1];
float zToCopy = pointsCoordinates[threadId * 3 + 2];
if(activityFlag[threadId] == 0)
{
xToCopy = xNonValid;
yToCopy = yNonValid;
zToCopy = zNonValid;
}
dataVertex[dataVertexOffset + threadId * 3] = xToCopy;
dataVertex[dataVertexOffset + threadId * 3 + 1] = yToCopy;
dataVertex[dataVertexOffset + threadId * 3 + 2] = zToCopy;
}
}