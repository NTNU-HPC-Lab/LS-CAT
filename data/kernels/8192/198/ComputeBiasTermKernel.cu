#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeBiasTermKernel(  float *biasTerm, float cFactor, float *winningFraction, int activeCells, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
biasTerm[threadId] = cFactor * ( 1.00f / activeCells - winningFraction[threadId]);
}
}