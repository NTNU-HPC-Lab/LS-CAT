#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RBMInputForwardAndStoreKernel( float *inputPtr, float *outputPtr, float *biasPtr, float *storePtr, bool applyBias, int thisLayerSize )
{
// i: current neuron id
int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (i < thisLayerSize)
{
float result = inputPtr[i];
if (applyBias)
result += biasPtr[i];
outputPtr[i] = result;
storePtr[i] = result;
}
}