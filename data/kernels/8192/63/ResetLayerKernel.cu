#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ResetLayerKernel( float *layer, float value, int count )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < count)
{
layer[threadId] = value;
}

}