#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExponentialFunctionKernel(float exponent, float* input, float* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (id < size)
{
output[id] = pow(input[id], exponent);
}
}