#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RBMCopyFilterKernel( float *weightPtr, float *filterPtr, int weightCount, int i, int thisLayerSize )
{

int weightIndex = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (weightIndex < weightCount)
{
filterPtr[weightIndex] = weightPtr[i + weightIndex * thisLayerSize];
}
}