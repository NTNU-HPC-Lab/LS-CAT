#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CumulateThroughTimeKernel(float* memoryBlocks, int count, int sequenceLength)
{
int memoryIdx = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (memoryIdx < count)
{
for (size_t i = 1; i < sequenceLength; i++)
{
int memoryBlockOffset = i * count;
memoryBlocks[memoryIdx] += memoryBlocks[memoryBlockOffset + memoryIdx];
}
}
}