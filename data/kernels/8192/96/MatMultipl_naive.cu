#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMultipl_naive (float * A, float * B, float * C , int nColsA , int nColsB , int sizeC ) {
int i_col = blockIdx.x * blockDim.x + threadIdx.x; /// index in row
int i_row = blockIdx.y * blockDim.y + threadIdx.y; /// index in column
int idx = i_row * nColsB + i_col;  // # of cols in B = # of cols in C
float Cvalue = 0;

if (idx < sizeC){
for (int e=0; e < nColsA; e++)
Cvalue += A[i_row * nColsA + e] * B[e * nColsB + i_col];
C[idx] = Cvalue;
}
}