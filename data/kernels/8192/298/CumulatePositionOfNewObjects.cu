#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CumulatePositionOfNewObjects(float* mask , float* maskNewIds , float* maskOut, int mask_size, int mask_cols, float* centers, int centers_size, int centers_columns){
int idx = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int icol = idx % mask_cols;
int irow = idx / mask_cols;

int i_mask, i_obj;

if (idx<mask_size){
i_mask = mask[idx];
i_obj  = maskNewIds[i_mask];
maskOut[idx] = i_obj;
if (i_obj*centers_columns+2<centers_size){
atomicAdd(centers + 0 + i_obj*centers_columns , (float)icol);
atomicAdd(centers + 1 + i_obj*centers_columns , (float)irow);
atomicAdd(centers + 2 + i_obj*centers_columns , 1.0f);
}
}
}