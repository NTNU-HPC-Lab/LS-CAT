#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyKernel(float* from, float* to, int size)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if(threadId < size)
{
to[threadId] = from[threadId];
}
}