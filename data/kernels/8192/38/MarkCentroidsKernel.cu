#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MarkCentroidsKernel( float *centroidCoordinates, float *visField, int imgWidth, int imgHeight, int centroids )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;
if(threadId < centroids)
{
int x = lrintf(centroidCoordinates[threadId * 2]);
int y = lrintf(centroidCoordinates[threadId * 2 + 1]);

visField[y * imgWidth + x] = -1.00f;

}
}