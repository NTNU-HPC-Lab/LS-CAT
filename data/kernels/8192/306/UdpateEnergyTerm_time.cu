#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UdpateEnergyTerm_time( float* energy, int energy_dim, int nPatches, float * idFocuser_focused , float par_time_increase_energy_on_focus, float par_time_decrease_energy_in_time)
{

int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int idDim   = id % energy_dim;
int idPatch = id / energy_dim;
if (id<energy_dim*nPatches){
if (idDim==0){ // time
if (idPatch==(int)(*idFocuser_focused)) // it is id that focuser just focused
energy[id] += par_time_increase_energy_on_focus;
else
energy[id] /= par_time_decrease_energy_in_time ;
}
}
}