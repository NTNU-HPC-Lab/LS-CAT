#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DecreaseErrorAndUtilityKernel(  float *localError, float *utility, int *activityFlag, int maxCells, float beta  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
if(activityFlag[threadId] == 1)
{
localError[threadId] -= beta * localError[threadId];
utility[threadId] -= beta * utility[threadId];
}
}
}