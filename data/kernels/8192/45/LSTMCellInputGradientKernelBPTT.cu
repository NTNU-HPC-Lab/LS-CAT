#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMCellInputGradientKernelBPTT( float *input, float *previousOutput,  float *cellInputDeltas, float *cellInputWeightGradient,  int inputCount, int previousOutputCount, int cellsPerBlock )
{
int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

int weightsPerCell = inputCount + previousOutputCount + 1;

if (weightId < weightsPerCell * previousOutputCount)
{
int fromId = weightId % weightsPerCell;
int toId = weightId / weightsPerCell;

int isFromInputUnit = fromId >= 0 && fromId < inputCount;
int isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
int isFromBiasUnit = fromId == (inputCount + previousOutputCount);

float inputFromWeight = isFromInputUnit * input[isFromInputUnit * fromId]
+ isFromPreviousOutputUnit * previousOutput[isFromPreviousOutputUnit * (fromId - inputCount)]
+ isFromBiasUnit * 1;

cellInputWeightGradient[weightId] = cellInputDeltas[toId] * inputFromWeight;
}
}