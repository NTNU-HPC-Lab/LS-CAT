#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NNResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;
int size =  outputWidth * outputHeight;

if (id < size)
{
int px = id % outputWidth;
int py = id / outputWidth;

float xRatio = (float)(inputWidth - 1) / (outputWidth);
float yRatio = (float)(inputHeight - 1) / (outputHeight);

int x = (int) (xRatio * (px+.5f));
int y = (int) (yRatio * (py+.5f));

output[py * outputWidth + px] = input[y*inputWidth + x];
}
}