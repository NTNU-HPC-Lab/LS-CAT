#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InterpolateFromMemBlock(float* input1, float* input2, float* output, float* weightMemBlock, int inputSize)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
if (weightMemBlock[0] <= 0)
{
output[threadId] = input1[threadId];
}
else if (weightMemBlock[0] >= 1)
{
output[threadId] = input2[threadId];
}
else
{
output[threadId] = (1 - weightMemBlock[0]) * input1[threadId] + weightMemBlock[0] * input2[threadId];
}
}
}