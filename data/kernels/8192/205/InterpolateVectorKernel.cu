#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InterpolateVectorKernel(  int r, int q, int f, int inputSize, float *referenceVector  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
referenceVector[r * inputSize + threadId] = 0.50f * (referenceVector[q * inputSize + threadId] + referenceVector[f * inputSize + threadId]);
}
}