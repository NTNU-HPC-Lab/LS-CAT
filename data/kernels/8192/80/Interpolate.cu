#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Interpolate(float* input1, float* input2, float* output, float weight, int inputSize)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < inputSize)
{
if (weight <= 0)
{
output[threadId] = input1[threadId];
}
else if (weight >= 1)
{
output[threadId] = input2[threadId];
}
else
{
output[threadId] = (1 - weight) * input1[threadId] + weight * input2[threadId];
}
}
}