#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void DecodeValues(float* superposition, int symbolSize, float* output, float* reliability, int numOfValues, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (threadId >= numOfValues)
return;


output[threadId] = 0;
reliability[threadId] = 0;

float* dir = threadId == 0 ? dirX : dirY;
float* negDir = threadId == 0 ? negDirX : negDirY;
float* origin = threadId == 0 ? originX : originY;

for (int i = 0; i < symbolSize; i++)
{
// output  = s.d - s.n = s.dir
// one of the values s.d or s.n will be (very close to) zero
output[threadId] += superposition[i] * dir[i] - superposition[i] * negDir[i];
// rel	   = s.o
reliability[threadId] += superposition[i] * origin[i];
}

// rel     = s.o + s.dir
reliability[threadId] += fabs(output[threadId]);
// output  = s.dir / (s.o + s.dir)
output[threadId] /= reliability[threadId];

// Since s = dir*t + o*(1-t) + noise, we get
// s.dir   = dir.dir*t + o.dir*(1-t) + dir.noise = t + 0 + dir.noise
// s.o     = o.dir*t   + o.o*(1-t)   + o.noise   = 0 + (1-t) + o.noise
// output  = t + dir.noise / (1 + dir.noise + o.noise)
// Note that dir.noise and o.noise should be very close to zero.
// This should make the decoding more precise when noise has similar dot product to dir and o.
}