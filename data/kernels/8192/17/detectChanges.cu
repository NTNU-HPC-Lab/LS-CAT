#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void detectChanges(float* a, float* b, float* result, int size, float value)
{
int threadId = blockDim.x*blockIdx.y*gridDim.x
+ blockDim.x*blockIdx.x
+ threadIdx.x;

if(threadId < size)
{
if(a[threadId] > b[threadId])
{
result[threadId] = value;
}
else if(a[threadId] <b[threadId])
{
result[threadId] = -value;
}
else
{
result[threadId] = 0;
}
}
}