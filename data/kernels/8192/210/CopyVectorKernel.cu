#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyVectorKernel(  float *from, int fromOffset, float *to, int toOffset, int vectorSize  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < vectorSize)
{
to[threadId + toOffset] = from[threadId + fromOffset];
}

}