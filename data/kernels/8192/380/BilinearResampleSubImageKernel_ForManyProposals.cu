#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleSubImageKernel_ForManyProposals(const float *input, float *output, const float* subImageDefs, bool safeBounds, int subImageDefsDim, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int numberSubImages, int outputSize)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

int px = id % outputWidth;  // line in the single output image
int subim_id = id / outputWidth / outputHeight;  // which image it is
int py = (id / outputWidth) % outputHeight;  // column in the single output image

if (id<outputSize)
{
float subImgCX = subImageDefs[0 + subim_id*subImageDefsDim]; // <-1, 1>
float subImgCY = subImageDefs[1 + subim_id*subImageDefsDim]; // <-1, 1>
float subImgDiameter = subImageDefs[2 + subim_id*subImageDefsDim]; // <0,1>

int maxDiameter = min(inputWidth - 1, inputHeight - 1);
int diameterPix = (int)(subImgDiameter * maxDiameter);

diameterPix = max(1, diameterPix);
diameterPix = min(maxDiameter, diameterPix);

int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

if (safeBounds)
{
subImgX = max(subImgX, 1);
subImgY = max(subImgY, 1);

subImgX = min(subImgX, inputWidth - diameterPix - 1);
subImgY = min(subImgY, inputHeight - diameterPix - 1);
}

float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

int x = (int) (xRatio * px);
int y = (int) (yRatio * py);

if (x + subImgX >= 0 && y + subImgY >= 0 &&
x + subImgX < inputWidth && y + subImgY < inputHeight)
{
//--- X and Y distance difference
float xDist = (xRatio * px) - x;
float yDist = (yRatio * py) - y;

//--- Points
float topLeft= input[(y + subImgY) * inputWidth + x + subImgX];
float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1 ];

float result =
topLeft * (1 - xDist) * (1 - yDist) +
topRight * xDist * (1 - yDist) +
bottomLeft * yDist * (1 - xDist) +
bottomRight * xDist * yDist;

output[py * outputWidth + px + subim_id*outputWidth*outputHeight] = result;
}
}
}