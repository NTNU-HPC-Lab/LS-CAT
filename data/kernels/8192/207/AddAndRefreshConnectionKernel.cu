#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddAndRefreshConnectionKernel(  int node1, int node2, int *activityFlag, int *connection, int *age, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < 1)
{
activityFlag[node1] = 1;
activityFlag[node2] = 1;

connection[node1 * maxCells + node2] = 1;
age[node1 * maxCells + node2] = 0;
connection[node2 * maxCells + node1] = 1;
age[node2 * maxCells + node1] = 0;
}
}