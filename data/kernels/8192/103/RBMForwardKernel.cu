#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activationProbability(float x, float sigma)
{
return 1.0 / (1.0 + expf(-sigma * x));
}
__global__ void RBMForwardKernel( float					*inputPtr, float					*outputPtr, float					*weightPtr, float					*biasPtr, float					sigma, int						prevLayerSize, int						thisLayerSize, bool					useDropoutMask, bool					useDropout, float					dropoutRate, float					*dropoutMask )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
// dropout this neuron
if (useDropout && useDropoutMask && !dropoutMask[j])
{
outputPtr[j] = 0;
}
else
{
float sum = 0.0;
int index = j;
for (i = 0; i < prevLayerSize; i++) {
sum += weightPtr[index] * inputPtr[i];
index += thisLayerSize;
}
// add bias
sum += biasPtr[j];

float result = activationProbability(sum, sigma);

// only used for reconstruction forward calls
if (useDropout && !useDropoutMask && dropoutRate < 1)
result /= dropoutRate;


// set output value
outputPtr[j] = result;
}


}
}