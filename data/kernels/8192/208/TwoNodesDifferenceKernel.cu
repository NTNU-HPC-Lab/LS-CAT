#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TwoNodesDifferenceKernel(  int nodeOne, int nodeTwo, int vectorLength, float *referenceVector, float *twoNodesDifference  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < vectorLength)
{
twoNodesDifference[threadId] = referenceVector[nodeOne * vectorLength + threadId] - referenceVector[nodeTwo * vectorLength + threadId];
}
}