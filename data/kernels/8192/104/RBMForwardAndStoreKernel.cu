#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activationProbability(float x, float sigma)
{
return 1.0 / (1.0 + expf(-sigma * x));
}
__global__ void RBMForwardAndStoreKernel( float					*inputPtr, float					*outputPtr, float					*weightPtr, float					*biasPtr, float					*storedOutputPtr, float					sigma, int						prevLayerSize, int						thisLayerSize, bool					useDropout, float					*dropoutMask )
{
// i: prev. layer neuron id
// j: current layer neuron id
int i;
int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (j < thisLayerSize)
{
// dropout this neuron
if (useDropout && !dropoutMask[j])
{
outputPtr[j] = 0;
storedOutputPtr[j] = 0;
}
else
{
float sum = 0.0;
int index = j;
for (i = 0; i < prevLayerSize; i++) {
sum += weightPtr[index] * inputPtr[i];
index += thisLayerSize;
}
// add bias
sum += biasPtr[j];

float result = activationProbability(sum, sigma);

// set output value
outputPtr[j] = result;

// store output value
storedOutputPtr[j] = result;
}
}
}