#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeDistanceKernel(  int inputSize, float *distance, float *dimensionWeight, int maxCells, float *difference  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells)
{
float sum = 0.00f;
float value;
for(int i = 0; i < inputSize; i++)
{
value = difference[threadId * inputSize + i];
sum += dimensionWeight[i] * value*value;
}
distance[threadId] = sqrtf(sum);
}
}