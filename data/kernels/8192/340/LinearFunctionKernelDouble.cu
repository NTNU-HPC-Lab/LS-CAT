#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearFunctionKernelDouble(double a1, double a0, double* input, double* output, int size)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if(id < size)
{
double x = input[id];
output[id] =  a1 * x + a0;
}
}