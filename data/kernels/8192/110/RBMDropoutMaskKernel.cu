#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RBMDropoutMaskKernel( float *maskPtr, float dropout, int thisLayerSize )
{

int index = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
+ blockDim.x * blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if (index < thisLayerSize)
{
maskPtr[index] = dropout < maskPtr[index];
}
}