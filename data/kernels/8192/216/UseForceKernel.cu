#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UseForceKernel(  float *force, float forceFactor, float *pointsCoordinates, int maxCells  )
{
int threadId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
+ blockDim.x*blockIdx.x				//blocks preceeding current block
+ threadIdx.x;

if(threadId < maxCells * 3)
{
pointsCoordinates[threadId] += forceFactor * force[threadId];
}
}