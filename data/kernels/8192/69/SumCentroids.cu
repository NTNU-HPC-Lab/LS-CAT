#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumCentroids(float* delta, float* sumDelta, int numOfCentroids, int numOfElements)
{
int id = blockDim.x * blockIdx.y * gridDim.x
+ blockDim.x * blockIdx.x
+ threadIdx.x;

if (id < numOfCentroids * NUM_SUMS)
{
float sum = 0;

for (int i = 0; i < numOfElements; i++)
{
sum += delta[numOfElements * id + i];
}

sumDelta[id] = sum;
}
}