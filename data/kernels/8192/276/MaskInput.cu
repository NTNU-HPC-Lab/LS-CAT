#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MaskInput( float* image, float* mask, float* maskedValues, float* output, int count ) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;

if (id < count) {
output[id] = image[id] * mask[id] + maskedValues[id] * (1.0f - mask[id]);
}
}