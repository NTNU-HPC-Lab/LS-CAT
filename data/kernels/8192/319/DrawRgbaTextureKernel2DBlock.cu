#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaTextureKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
int id = blockDim.x * blockDim.y * (blockIdx.y * gridDim.x + blockIdx.x)
+ blockDim.x * threadIdx.y
+ threadIdx.x; // 2D grid of 2D blocks; block dimension x = texture width;
// grid dimension x + block dimension y = texture height

int targetPixels = targetWidth * targetHeight;

int texturePixels = textureWidth * textureHeight;

int idTextureRgb = blockIdx.y;
int idTexturePixel = (id - idTextureRgb * texturePixels);
int idTextureY = blockIdx.x * blockDim.y + threadIdx.y;
int idTextureX = threadIdx.x;


if (idTextureRgb < 3) // 3 channels that we will write to
{
// the texture is in BGR format, we want RGB
switch (idTextureRgb)
{
case 0: // R
idTextureRgb = 2; // B
break;
case 2: // B
idTextureRgb = 0; // R
break;
}
// if the texture pixel offset by inputX, inputY, lies inside the target
if (idTextureX + inputX < targetWidth &&
idTextureX + inputX >= 0 &&
idTextureY + inputY < targetHeight &&
idTextureY + inputY >= 0)
{
int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
float a = texture[aIndex];
target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
}
}
}