#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UdpateEnergyTerm_movement( float* energy, int energy_dim, int nPatches, float * desc, int desc_dim, int id_desc_move) // whic hindex is the one with movement
{

int id = blockDim.x*blockIdx.y*gridDim.x	+ blockDim.x*blockIdx.x	+ threadIdx.x;
int idDim   = id % energy_dim;
int idPatch = id / energy_dim;
if (id<energy_dim*nPatches){
if (idDim==1) // movement
energy[id] = -desc[idPatch*desc_dim + id_desc_move];
}
}