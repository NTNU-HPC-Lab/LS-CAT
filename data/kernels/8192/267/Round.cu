#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Round(float * A, float  *out, int size) {
int id = blockDim.x*blockIdx.y*gridDim.x + blockDim.x*blockIdx.x + threadIdx.x;
if (id < size) {
int t = (int)(out[id] + 0.5);  // can it be speeded up??
out[id] = (float)t;
}
}