#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _setPrecisionKernel(float* data, size_t size, int* precision)
{
unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx >= size) return;
int prec = precision[idx];
int mul = 1;
while(prec--) mul *= 10;
data[idx] = (float)(int)(data[idx]*mul);
data[idx] /= mul;
}