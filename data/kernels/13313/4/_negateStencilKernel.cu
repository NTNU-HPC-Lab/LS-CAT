#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _negateStencilKernel(int* stencil, int size, int* out)
{
unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx >= size) return;

out[idx] = stencil[idx] == 1 ? 0 : 1;
}