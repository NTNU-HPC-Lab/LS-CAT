#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blur(unsigned char *pixels, int rows, int cols, int channels, int kernel, int numThreads){
int id = blockDim.x * blockIdx.x + threadIdx.x ;
int i = rows * id / numThreads;
int end = ( id == numThreads-1)? rows: rows * (id + 1) / numThreads;

int k = kernel/2;

for(; i<end; i++){
for (int j=0; j<cols; j++){
unsigned int blue=0.0, red=0.0, green=0.0;
double sum = 0.0;
for(int x=i-k; x<=i+k; x++){
for(int y=j-k; y<=j+k; y++){
if(x<rows && x>=0 && y<cols && y>=0){
sum += 1;
blue  += pixels[ (cols*x*channels+y*channels)+0];
green += pixels[ (cols*x*channels+y*channels)+1];
red   += pixels[ (cols*x*channels+y*channels)+2];
}
}
}
pixels[ (cols*i*channels+j*channels)+0] = (unsigned int) blue/sum;
pixels[ (cols*i*channels+j*channels)+1] = (unsigned int) green/sum;
pixels[ (cols*i*channels+j*channels)+2] = (unsigned int) red/sum;
}
}
}