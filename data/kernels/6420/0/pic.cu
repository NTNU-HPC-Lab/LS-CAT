#include "hip/hip_runtime.h"
#include "includes.h"

#define ITER  10000000000  // Number of bins
#define NUMBLOCKS   13  // Number of thread blocks
#define NUMTHREADS 192  // Number of threads per block
int tid;
float pi;

// Kernel

// Main
__global__ void pic(float *sum, int nbin, float step, int nthreads, int nblocks) {
int i;
float x;
int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
for (i=idx; i< nbin; i+=nthreads*nblocks) {
x = (i+0.5)*step;
sum[idx] += 4.0/(1.0+x*x);
}
}