#include "hip/hip_runtime.h"
#include "includes.h"
// ïîäêëþ÷åíèå áèáëèîòåêè cuBLAS


#define IDX2C(i,j,ld) (((i)*(ld))+(j))


__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N)
{
int ROW = blockIdx.y*blockDim.y + threadIdx.y;
int COL = blockIdx.x*blockDim.x + threadIdx.x;
float tmpSum = 0;

if (ROW < N && COL < N) {
// each thread computes one element of the block sub-matrix
for (int i = 0; i < N; i++) {
tmpSum += A[ROW * N + i] * B[i * N + COL];
}

C[ROW * N + COL] = tmpSum;
}
}