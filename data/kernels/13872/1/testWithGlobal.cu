#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float compute(int idx, float* buf, int s)
{
// some random calcs to make the kernel unempty
float k=0.0f;
for (int x=0;x<s;x++ ){
k+=cosf(x*0.1f*idx);
buf[x]=k;
}
for (int x=0;x<s/2;x++){
buf[x]=buf[x]*buf[x];
}
float sum=0.0f;
for (int x=s-1;x>=1;x--) {
sum += buf[x-1]/(fabsf(buf[x])+0.1f);
}
return sum;
}
__global__ void testWithGlobal(int n, int s, float* result, float* buf) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < n) {
result [idx] = compute(idx, &buf [idx * s],s);
}
}