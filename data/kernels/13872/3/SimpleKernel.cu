#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SimpleKernel(int N, float* a){
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < N) {
for (int x=0;x<1000;x++)
a[idx] = asin(a[idx]+x);
}
}