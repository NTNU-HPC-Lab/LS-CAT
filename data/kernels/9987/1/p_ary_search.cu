#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {

// Integer division trick to round up
return (((set_size + num_threads) / num_threads) * thread) + offset;
}
__global__ void p_ary_search(int search, int array_length, int *arr, int *ret_val)
{
const int num_threads = blockDim.x * gridDim.x;
const int thread = blockIdx.x * blockDim.x + threadIdx.x;
int set_size = array_length;

ret_val[0] = -1;
ret_val[1] = 0;

while (set_size != 0)
{
int offset = ret_val[1];

__syncthreads();

// Get the next index to check
int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

// If the index is outside the bounds of the array do not check it
if (index_to_check < array_length)
{
// If the next index is outside the bounds of the array, then set it to maximum array size
int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);
if (next_index_to_check >= array_length)
{
next_index_to_check = array_length - 1;
}

// If we're at the mid section of the array reset the offset to this index
if (search > arr[index_to_check] && (search < arr[next_index_to_check]))
{
ret_val[1] = index_to_check;
}
else if (search == arr[index_to_check])
{
// Set the return var if find it
ret_val[0] = index_to_check;
}
}

// Since this is a paralel array search divide by our total threads to get the next set size
set_size = set_size / num_threads;

// Sync up so no threads jump ahead
__syncthreads();
}
}