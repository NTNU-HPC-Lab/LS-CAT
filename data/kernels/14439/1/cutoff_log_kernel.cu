#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void cutoff_log_kernel(double* device_array, double min_signal){
int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
if (device_array[thread_id] < min_signal){
device_array[thread_id] = logf(min_signal);
}
else{
device_array[thread_id] = logf(device_array[thread_id]);
}
}