#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void transpose_kernel(double const* matrices, double* transposed) {
int matrix_offset = blockIdx.x * blockDim.x * blockDim.y;
int matrix_index = matrix_offset + blockDim.x * threadIdx.y + threadIdx.x;
int transpose_index = matrix_offset + IDX2C(threadIdx.y, threadIdx.x, blockDim.y);
transposed[transpose_index] = matrices[matrix_index];
}