#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void multiply_arrays(double* signals, double const* weights){
signals[blockIdx.x * blockDim.x + threadIdx.x] *= weights[blockIdx.x * blockDim.x + threadIdx.x];
}