#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void weighting_kernel (double const* matrices, double const* weights, double* results) {
int matrix_grid_index = blockIdx.x * blockDim.x * blockDim.y;
int block_index = blockDim.y * threadIdx.x + threadIdx.y;
int matrix_index = matrix_grid_index + block_index;
int weight_index = blockIdx.x * blockDim.y + threadIdx.y;
results[matrix_index] = matrices[block_index] * weights[weight_index];
}