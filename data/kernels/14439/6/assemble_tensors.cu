#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void assemble_tensors(double const* tensor_input, double* tensors, int tensor_input_elements){
int tensor_matrix_offset = blockIdx.x * TENSOR_DIMENSIONS * TENSOR_DIMENSIONS;
int input_matrix_offset = blockIdx.x * tensor_input_elements;
tensors[tensor_matrix_offset + 0] = tensor_input[input_matrix_offset + 0];
tensors[tensor_matrix_offset + 1] = tensor_input[input_matrix_offset + 1];
tensors[tensor_matrix_offset + 2] = tensor_input[input_matrix_offset + 3];
tensors[tensor_matrix_offset + 3] = tensor_input[input_matrix_offset + 1];
tensors[tensor_matrix_offset + 4] = tensor_input[input_matrix_offset + 2];
tensors[tensor_matrix_offset + 5] = tensor_input[input_matrix_offset + 4];
tensors[tensor_matrix_offset + 6] = tensor_input[input_matrix_offset + 3];
tensors[tensor_matrix_offset + 7] = tensor_input[input_matrix_offset + 4];
tensors[tensor_matrix_offset + 8] = tensor_input[input_matrix_offset + 5];
}