#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void weighting_kernel_transposed(double const* matrices, double const* weights, double* results) {
int grid_index = blockIdx.x * blockDim.x * blockDim.y;
int block_index = blockDim.y * threadIdx.x + threadIdx.y;
int matrix_index = grid_index + block_index;
int weighting_index = blockIdx.x * blockDim.x + threadIdx.x;
results[matrix_index] = matrices[block_index] * weights[weighting_index];
}