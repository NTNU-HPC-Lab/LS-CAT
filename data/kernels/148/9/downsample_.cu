#include "hip/hip_runtime.h"
#include "includes.h"


#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void downsample_(float *input, float *output, int factor, int size3, int size)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
int dim3 = id % size3;
int dim2 = id / size3;
atomicAdd(output + ((dim2 / factor) * (size3 / factor) + (dim3 / factor)), input[id] / (factor * factor));
}
}