#include "hip/hip_runtime.h"
#include "includes.h"


#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void Normalize_forward_(float *input, float *norm, float *output, int size23, int size123, int size0123)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size0123) {
int dim23 = id % size23;
int dim0 = (id / size123);
output[id] = input[id] / sqrtf(norm[dim0 * size23 + dim23]);
}
}