#include "hip/hip_runtime.h"
#include "includes.h"


#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void fill_(float *input, float value, int size)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
input[id] = value;
}
}