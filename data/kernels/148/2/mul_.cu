#include "hip/hip_runtime.h"
#include "includes.h"


#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void mul_(float *input, float factor, int size)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size) {
input[id] = input[id] * factor;
}
}