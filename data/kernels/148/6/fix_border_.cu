#include "hip/hip_runtime.h"
#include "includes.h"


#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void fix_border_(float *input, int pad_size, int side, int size3, int size23)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < size23) {
int x = id % size3;
int y = id / size3;
if (side == 0 && x < pad_size) {
input[id] = input[y * size3 + pad_size];
} else if (side == 1 && x > size3 - pad_size - 1) {
input[id] = input[y * size3 + size3 - pad_size - 1];
}
}
}