#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hello_from_gpu(void)
{
int bid = blockIdx.x;
int tid = threadIdx.x;
printf("Hello World from block %d and thread %d!\n", bid, tid);
}