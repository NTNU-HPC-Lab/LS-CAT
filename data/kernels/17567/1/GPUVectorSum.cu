#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPUVectorSum(int * a, int * b, int * c, int VECTOR_QNT) {
int n = VECTOR_QNT;
int idx = blockIdx.x * blockDim.x + threadIdx.x;
for (int i = idx; i < n; i += blockDim.x * gridDim.x)
{
c[i] = a[i] + b[i];
}
}