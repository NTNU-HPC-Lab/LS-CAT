#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_regularize_kernel(int factors, float regularization, float * YtY) {
YtY[threadIdx.x * factors + threadIdx.x] += regularization;
}