#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc(float *d_D, int n, int k){ //kernel (4  cells for every thread)
__shared__ float s_d[4*3*256]; //Shared table within a block
int i = blockIdx.x * blockDim.x + threadIdx.x; //Calculation of i and j
int j = blockIdx.y * blockDim.y + threadIdx.y;
int b_index = 4 * 3 * (threadIdx.x + blockDim.x*threadIdx.y); //Calculation of initial index of thread in the shared table within the block
int istep = blockDim.x*gridDim.x, jstep = blockDim.y*gridDim.y;
int l, m , v=0;
for (l = 0; l<2; l++){
for (m = 0; m<2; m++){ //Pass values from device table to shared block table for every one of the 4 cells
s_d[b_index + 3 * v] = d_D[(i+l*istep)+(j+m*jstep)*n];
s_d[b_index + (3 * v + 1)] = d_D[(i + l*istep) + k*n];
s_d[b_index + (3 * v + 2)] = d_D[k + (j + m*jstep)*n];
v++;
}
}
for (v = 0; v<4; v++){ //Calculate the new cell values (4 for every thread)
if (s_d[b_index + 3 * v] > s_d[b_index + (3 * v + 1)] + s_d[b_index + (3 * v + 2)]) s_d[b_index + 3 * v] = s_d[b_index + (3 * v + 1)] + s_d[b_index + (3 * v + 2)];
}
v = 0;
for (l = 0; l<2; l++){ //Pass the new values to the device table
for (m = 0; m<2; m++){
d_D[(i+l*istep)+(j+m*jstep)*n] = s_d[b_index + 3 * v];
v++;
}
}
}