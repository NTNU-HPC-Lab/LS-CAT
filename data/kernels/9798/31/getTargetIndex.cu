#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/


extern "C"

extern "C"


extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

__global__ void getTargetIndex(int n, int *index, double *w)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
if(w[i] == 1.0) {index[0] = i;}
}
}