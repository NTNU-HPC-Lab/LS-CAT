#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/


extern "C"

extern "C"


extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

__global__ void mismatch(int n, double* actual, double *target, int *mis)
{

mis[0] = 0;

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
if(target[i] >= 0.5 && actual[i] < 0.5) {mis[0] = 1;}
if(target[i] < 0.5 && actual[i] >= 0.5) {mis[0] = 1;}
}
}