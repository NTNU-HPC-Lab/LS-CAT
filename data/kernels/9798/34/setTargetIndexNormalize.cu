#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/


extern "C"

extern "C"


extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

__global__ void setTargetIndexNormalize(int n, double sum, double *w, double *out, double* output)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
if(w[i] == 1.0) {output[0] = out[i]/sum;}
}
}