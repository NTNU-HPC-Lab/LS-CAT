#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/


extern "C"

extern "C"


extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

__global__ void backwardError(int n, double *actual, double *target, double* out)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i<n)
{
out[i] += (actual[i] - target[i]);
}
}