#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/


extern "C"

extern "C"


extern "C"


extern "C"

extern "C"

extern "C"

extern "C"

__global__ void normalize(double *g_idata, double *g_odata, unsigned int n, int maxIndx)
{

double max = g_idata[maxIndx];
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < n)
{
g_odata[i] = exp(g_idata[i] - max);
}

}