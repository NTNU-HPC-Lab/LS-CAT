#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  convolutionGPUkernel_2D(int *M, int *mascara,int *resultado,int m, int n, int widthM){
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

if(col < n && row < m){
int p = 0;
int start_col = col - (widthM/2);
int start_row = row - (widthM/2);
for (int i = 0; i < widthM ; i++) {
for (int j = 0; j < widthM; j++) {
int curRow = start_row + i;
int curCol = start_col + j;
if(curRow > -1 && curRow < m && curCol > -1 && curCol < n){
p += M[curRow*m + curCol]*mascara[i*widthM + j];
}
}
}
resultado[row*n + col] = p;
}
}