#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  convolutionGPUkernel_1D(int *h_n, int *h_mascara,int *h_r,int n, int mascara){
int mitadMascara= (mascara/2);
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i<n){
int p=0;// almacena los valores temporales
int k= i - mitadMascara;
for (int j =0; j < mascara; j++){
if(k < n  && k >= 0){
p += h_n[k]*h_mascara[j];
}
else
p+=0;
k++;
}
h_r[i]=p;
}
}