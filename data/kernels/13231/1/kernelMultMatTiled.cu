#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMultMatTiled(float *d_M, float *d_N, float *d_P, int m,int n , int y){


// se define la memoria compartida de los tiles de tamaño TILE_WIDTH

__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;
int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;
float Pvalue = 0;

for(int i = 0; i < n / TILE_WIDTH; i++){
/* primeramente se revisa que el elemento se encuentre en la matriz d_M ,
si no es así se establecen como cero
*/
if((i*TILE_WIDTH + tx) < n && row < m){
Mds[ty][tx]=d_M[row*n + (i*TILE_WIDTH + tx)];
}else{
Mds[ty][tx]=0.0;
}
/* despues  se revisa que el elemento se encuentre en la matriz d_N ,
si no es así se establecen como cero
*/
if((i*TILE_WIDTH + ty) < n && col < y){
Nds[ty][tx]= d_N[(i*TILE_WIDTH + ty)*y + col];
}else{
Nds[ty][tx]=0.0;
}
__syncthreads();
/*Se realiza la multiplicacion de elementos que están dentro del TILE
y se va guardando en Pvalue*/
for(int k = 0; k < TILE_WIDTH; ++k){
Pvalue += Mds[ty][k] * Nds[k][tx];
}
__syncthreads();
}
//se asigna el resultado de Pvalue en las posiciones de d_P
if(row<m && col < y)
d_P[(row*y)+ col] = Pvalue;
}