#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMultMat(double *d_a, double *d_b, double *d_c, int ROWS, int COL_A, int COL_B) {

int col = blockIdx.x * blockDim.x + threadIdx.x;
int row = blockIdx.y * blockDim.y + threadIdx.y;
double add;

if (row < ROWS && col < COL_B) {
add = 0;
for (int k = 0; k < COL_A; k++) {
add += d_a[row * COL_A + k] * d_b[k * COL_B + col];
}
d_c[row * COL_B + col] = add;
}
}