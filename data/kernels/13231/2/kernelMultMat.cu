#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMultMat(int *a, int *b, int *c,int m){
int i,add;

int col=blockDim.x*blockIdx.x + threadIdx.x;
int row=blockDim.y*blockIdx.y + threadIdx.y;

if(col<m && row<m) {
add=0;
for(i=0; i< m ;i++){
add += a[i+m*row]*b[col+m*i];
}
c[row*m+col] = add;
}
}