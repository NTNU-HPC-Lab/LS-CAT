#include "hip/hip_runtime.h"
#include "includes.h"



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
c[i] = a[i] + b[i];
}