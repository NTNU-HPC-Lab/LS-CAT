#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultiSoftmaxCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, float* top5Probs, const int numCases, const int numOut, const int setSize) {
const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

if (tx < numCases) {
const int label = int(labels[tx]);
const float maxp = maxProbs[tx];
const float labelp = probs[label * numCases + tx];

labelLogProbs[tx] = __logf(labelp);

int numBiggerProbs = 0, numEqualsProbs = 0;
for (int i = 0; i < numOut; ++i) {
numBiggerProbs += probs[i * numCases + tx] > labelp;
numEqualsProbs += probs[i * numCases + tx] == labelp;
}

const int slotsLeft = setSize - numBiggerProbs;

top5Probs[tx] = slotsLeft <= 0.0f ? 0.0f : (numEqualsProbs <= slotsLeft ? 1.0f : float(slotsLeft) / numEqualsProbs);
correctProbs[tx] = labelp != maxp ? 0.0f : 1.0f / float(numEqualsProbs);
}
}