#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
float pixel = 0.0f;

for (int h = 0; h < filter_dim; h++)
{
int offset        = h * stride;
int offset_kernel = h * filter_dim;

for (int w = 0; w < filter_dim; w++)
{
pixel += image[offset + w] * matrix[offset_kernel + w];
}
}

return pixel;
}
__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
////////////////
// TO-DO #6.1 /////////////////////////////////////
// Implement the GPU version of the Sobel filter //
///////////////////////////////////////////////////
__shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];

float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
2.0f,  0.0f, -2.0f,
1.0f,  0.0f, -1.0f };
float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
0.0f,  0.0f,  0.0f,
-1.0f, -2.0f, -1.0f };

int index_x = blockIdx.x * blockDim.x + threadIdx.x;
int index_y = blockIdx.y * blockDim.y + threadIdx.y;

int offset_t = index_y * width + index_x; // Input for function
int offset   = (index_y + 1) * width + (index_x + 1); // Output to store in result

// Shared memory offset (for input value):
int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

if (index_x != 0 && (index_x+1) % BLOCK_SIZE == 0) {
// Edge-case x-direction:
sh_block[offset_shared + 1] = image[offset_t + 1];
sh_block[offset_shared + 2] = image[offset_t + 2];
}
if (index_y != 0 && (index_y+1) % BLOCK_SIZE == 0) {
// Edge-case y-direction:
sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
sh_block[offset_shared + 2*BLOCK_SIZE_SH] = image[offset_t + 2*width];
}
if ((index_x != 0 && (index_x+1) % BLOCK_SIZE == 0) && (index_y != 0 && (index_y+1) % BLOCK_SIZE == 0)) {
// Edge-case x & y-direction:
sh_block[offset_shared + BLOCK_SIZE_SH + 1] = image[offset_t + width + 1];
sh_block[offset_shared + BLOCK_SIZE_SH + 2] = image[offset_t + width + 2];
sh_block[offset_shared + 2*BLOCK_SIZE_SH + 1] = image[offset_t + 2*width + 1];
sh_block[offset_shared + 2*BLOCK_SIZE_SH + 2] = image[offset_t + 2*width + 2];
}

sh_block[offset_shared] = image[offset_t];
__syncthreads();

if (index_x < (width - 2) && index_y < (height - 2))
{
float gx = cpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_x, 3);
float gy = cpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_y, 3);
image_out[offset] = sqrtf(gx * gx + gy * gy);
}

}