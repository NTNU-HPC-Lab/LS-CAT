#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(float *x, float *y, const float a)
{

const int i = blockIdx.x*blockDim.x + threadIdx.x;

if (i<ARRAY_SIZE) {
y[i] = a*x[i] + y[i];
}
}