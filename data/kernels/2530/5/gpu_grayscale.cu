#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
////////////////
// TO-DO #4.2 /////////////////////////////////////////////
// Implement the GPU version of the grayscale conversion //
///////////////////////////////////////////////////////////

const int x = blockIdx.x*blockDim.x + threadIdx.x;
const int y = blockIdx.y*blockDim.y + threadIdx.y;

if (x < width && y < height) {

int offset_out = ((width * y) + x);
int offset = offset_out*3;

float *pixel = &image[offset];

image_out[offset_out] = pixel[0] * 0.0722f + // B
pixel[1] * 0.7152f + // G
pixel[2] * 0.2126f;  // R

}
}