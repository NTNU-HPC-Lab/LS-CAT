#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
float pixel = 0.0f;

for (int h = 0; h < filter_dim; h++)
{
int offset        = h * stride;
int offset_kernel = h * filter_dim;

for (int w = 0; w < filter_dim; w++)
{
pixel += image[offset + w] * matrix[offset_kernel + w];
}
}

return pixel;
}
__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{
float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };

int index_x = blockIdx.x * blockDim.x + threadIdx.x;
int index_y = blockIdx.y * blockDim.y + threadIdx.y;

int offset_t = index_y * width + index_x; // Input for function
int offset   = (index_y + 1) * width + (index_x + 1); // Output to store in result

// sh_block = handle_shared_memory(image, offset_t, index_x, index_y, width);

__shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];

// Shared memory offset (for input value):
int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

if (index_x != 0 && (index_x+1) % BLOCK_SIZE == 0) {
// Edge-case x-direction:
sh_block[offset_shared + 1] = image[offset_t + 1];
sh_block[offset_shared + 2] = image[offset_t + 2];
}
if (index_y != 0 && (index_y+1) % BLOCK_SIZE == 0) {
// Edge-case y-direction:
sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
sh_block[offset_shared + 2*BLOCK_SIZE_SH] = image[offset_t + 2*width];
}
if ((index_x != 0 && (index_x+1) % BLOCK_SIZE == 0) && (index_y != 0 && (index_y+1) % BLOCK_SIZE == 0)) {
// Edge-case x & y-direction:
sh_block[offset_shared + BLOCK_SIZE_SH + 1] = image[offset_t + width + 1];
sh_block[offset_shared + BLOCK_SIZE_SH + 2] = image[offset_t + width + 2];
sh_block[offset_shared + 2*BLOCK_SIZE_SH + 1] = image[offset_t + 2*width + 1];
sh_block[offset_shared + 2*BLOCK_SIZE_SH + 2] = image[offset_t + 2*width + 2];
}

sh_block[offset_shared] = image[offset_t];
__syncthreads();

if (index_x < (width - 2) && index_y < (height - 2))
{
image_out[offset] = cpu_applyFilter(&sh_block[offset_shared],
BLOCK_SIZE_SH, gaussian, 3);
}
}