#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_kernel_texture_2d(unsigned char *surface, int width, int height, size_t pitch, float t) {
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

// in the case where, due to quantization into grids, we have
// more threads than pixels, skip the threads which don't
// correspond to valid pixels
if (x >= width || y >= height) {
return;
}

// get a pointer to the pixel at (x,y)
float *pixel = (float *)(surface + y * pitch) + 4 * x;

// populate it
float value_x = 0.5f + 0.5f * cos(t + 10.0f * ((2.0f * x) / width  - 1.0f));
float value_y = 0.5f + 0.5f * cos(t + 10.0f * ((2.0f * y) / height - 1.0f));

pixel[0] = 0.5 * pixel[0] + 0.5 * pow(value_x, 3.0f); // red
pixel[1] = 0.5 * pixel[1] + 0.5 * pow(value_y, 3.0f); // green
pixel[2] = 0.5f + 0.5f * cos(t); // blue
pixel[3] = 1.0f; // alpha
}