#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void threshold_and_support(float *vec, int *support, const int n, const float T)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if (xIndex < n) {
if (abs(vec[xIndex])<T) {
vec[xIndex] = 0.0f;
support[xIndex]=2;
}
}
}