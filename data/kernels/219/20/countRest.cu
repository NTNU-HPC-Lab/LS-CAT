#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void countRest(int *bin, int *bin_counters, const int num_bins, const int maxBin, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( (xIndex < n) & (bin[xIndex]<num_bins) )
if (bin[xIndex]>= maxBin) atomicAdd(bin_counters+bin[xIndex],1);
}