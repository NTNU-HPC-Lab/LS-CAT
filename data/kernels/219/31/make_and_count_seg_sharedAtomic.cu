#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void MakeCountSegment_sharedAtomic(float *segment, int *bins, const int seglength, int *segCounter, int *s_counter, const int countlength, const float low, const float high, const float slope)
{
int bin;
float temp;
for (int jj=0; jj<seglength; jj++){
temp = abs(segment[jj]);
if ( ( temp > low ) & ( temp < high ) ) {
bin = (int)ceil(slope*abs(high-temp));
}
else if (temp >= high) {
bin = 0;
}
else bin = countlength - 1;
bins[jj]=bin;
atomicAdd(s_counter+bin,1);
}

for (int jj=0; jj<countlength; jj++) segCounter[jj]=s_counter[jj];

return;
}
__device__ void MakeCountSegment(float *segment, int *bins, const int seglength, int *segCounter, const int countlength, const float low, const float high, const float slope)
{
int bin;
float temp;
for (int jj=0; jj<seglength; jj++){
temp = abs(segment[jj]);
if ( ( temp > low ) & ( temp < high ) ) {
bin = (int)ceil(slope*abs(high-temp));
}
else if (temp >= high) {
bin = 0;
}
else bin = countlength - 1;
bins[jj]=bin;
segCounter[bin] = segCounter[bin] + 1;
}

return;
}
__global__ void make_and_count_seg_sharedAtomic(float *vec, int *bin, int *segcounter, const int length, const int countlength, const int HighLength, const int HighSegmentLength, const int threadsHigh, const int LowSegmentLength, const float low, const float high, const float slope)
{
int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
int startIndex, SegmentLength, startCountIndex;
extern __shared__ int s_counter[];

startCountIndex = xIndex*countlength;

if ( (xIndex*HighSegmentLength > HighLength) & ( (HighLength + (xIndex-threadsHigh+1)*LowSegmentLength) < length ) ){
startIndex = HighLength + (xIndex-threadsHigh)*LowSegmentLength;
SegmentLength = LowSegmentLength;
}
else {
startIndex = xIndex*HighSegmentLength;
SegmentLength = HighSegmentLength;
}
MakeCountSegment_sharedAtomic(vec+startIndex, bin+startIndex, SegmentLength, segcounter+startCountIndex, s_counter, countlength, low, high, slope);
}