#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void one_vector_float(float *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < n )
vec[xIndex]=1.0f;
}