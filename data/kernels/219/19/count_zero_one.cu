#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_zero_one(float *vec, float *data, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( (xIndex < n) ){
if (vec[xIndex] == 0)
atomicAdd(data,1);
else if (vec[xIndex] == 1)
atomicAdd(data+1,1);
}
}