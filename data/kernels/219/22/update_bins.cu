#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_bins(float *vec, int *bin, int *bin_counters, const int num_bins, const int n, const float slope, const float intercept)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if ( xIndex < n ){
int bin_new_val;
float temp = abs(vec[xIndex]);
if ( temp > (intercept *.000001) ){
bin_new_val=slope * (intercept - temp);
}
else bin_new_val = num_bins;

if ( bin[xIndex] != bin_new_val ){
if (bin[xIndex] < num_bins)
atomicAdd(bin_counters+bin[xIndex],-1);
if ( bin_new_val < num_bins )
atomicAdd(bin_counters+bin[xIndex],1);
bin[xIndex]=bin_new_val;
}


}
}