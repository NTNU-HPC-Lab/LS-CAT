#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_adjacent(int *vec, int *vec_shorter, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if ( xIndex < n )
vec_shorter[xIndex] = vec[2 * xIndex] + vec[(2 * xIndex) +1];

}