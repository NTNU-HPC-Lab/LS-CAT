#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearBinning(float *vec, int *bin, int *bin_counters, const int num_bins, const int MaxBin, const int n, const float slope, const float intercept)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
float temp = abs(vec[xIndex]);
if ( xIndex < n ){
if ( temp > (intercept *.000001) ){
bin[xIndex]=max(0.0f,slope * (intercept - temp));
if (bin[xIndex]<MaxBin) atomicAdd(bin_counters+bin[xIndex],1);
}
else bin[xIndex] = slope * intercept + 1.0f;
}
}