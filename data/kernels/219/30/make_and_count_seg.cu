#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void MakeCountSegment(float *segment, int *bins, const int seglength, int *segCounter, const int countlength, const float low, const float high, const float slope)
{
int bin;
float temp;
for (int jj=0; jj<seglength; jj++){
temp = abs(segment[jj]);
if ( ( temp > low ) & ( temp < high ) ) {
bin = (int)ceil(slope*abs(high-temp));
}
else if (temp >= high) {
bin = 0;
}
else bin = countlength - 1;
bins[jj]=bin;
segCounter[bin] = segCounter[bin] + 1;
}

return;
}
__global__ void make_and_count_seg(float *vec, int *bin, int *segcounter, const int length, const int countlength, const int HighLength, const int HighSegmentLength, const int threadsHigh, const int LowSegmentLength, const float low, const float high, const float slope)
{
int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
int startIndex, SegmentLength, startCountIndex;

startCountIndex = xIndex*countlength;

if ( (xIndex*HighSegmentLength > HighLength) & ( (HighLength + (xIndex-threadsHigh+1)*LowSegmentLength) < length ) ){
startIndex = HighLength + (xIndex-threadsHigh)*LowSegmentLength;
SegmentLength = LowSegmentLength;
}
else {
startIndex = xIndex*HighSegmentLength;
SegmentLength = HighSegmentLength;
}
MakeCountSegment(vec+startIndex, bin+startIndex, SegmentLength, segcounter+startCountIndex, countlength, low, high, slope);
}