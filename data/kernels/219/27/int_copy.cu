#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void int_copy(int *vec_to, int *vec_from, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if ( xIndex < n )
vec_to[xIndex] = vec_from[xIndex];

}