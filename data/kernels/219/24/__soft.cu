#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __soft(float* y, const float* x, float T, int m)
{

unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
float x_e, y_e;

if(xIndex < m)
{
x_e = x[xIndex];
y_e = fmaxf(fabsf(x_e) - T, 0.f);
y[xIndex] = y_e / (y_e + T) * x_e;
}
}