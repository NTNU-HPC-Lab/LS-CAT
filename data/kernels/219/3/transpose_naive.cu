#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_naive(float *odata, float* idata, int width, int height)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

if (xIndex < width && yIndex < height)
{
unsigned int index_in  = xIndex + width * yIndex;
unsigned int index_out = yIndex + height * xIndex;
odata[index_out] = idata[index_in];
}
}