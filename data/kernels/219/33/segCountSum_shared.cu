#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void segCountSum_shared(int *counter, int *segcounter, const int countlength)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
extern __shared__ int s_counter[];

if (xIndex < countlength){
for (int jj=0; jj<countlength; jj++){
s_counter[xIndex] = s_counter[xIndex] + segcounter[xIndex + jj*countlength];
}
}
counter[xIndex] = s_counter[xIndex];
}