#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dyadicAdd(int * counter, const int length, const int shift)
{
if (shift > 0) {
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
int adds = 2*shift;
int Index = adds*(xIndex+1)-1;

if (Index < length) {
counter[Index] = counter[Index] + counter[Index-shift];
}
}
}