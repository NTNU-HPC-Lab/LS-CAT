#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void threshold_one(float *vec, float *vec_thres, int *bin, const int k_bin, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
// xIndex is a value from 1 to k from the vector ind

if ( (xIndex < n) & (bin[xIndex]<=k_bin) )
vec_thres[xIndex]=vec[xIndex];
}