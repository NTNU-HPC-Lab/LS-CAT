#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count_bins(int *bin, int *bin_counters, const int num_bins, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( (xIndex < n) & (bin[xIndex]<num_bins) )
atomicAdd(bin_counters+bin[xIndex],1);
}