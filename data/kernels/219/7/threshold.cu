#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void threshold(float *vec, int *bin, const int k_bin, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
// xIndex is a value from 1 to k from the vector ind

if ( (xIndex < n) & (bin[xIndex]>k_bin) )
vec[xIndex]=0.0f;
}