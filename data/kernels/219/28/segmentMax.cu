#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float getAbsMax(float * d_vec, const int length)
{
int jj=0;
float segmentMax = 0;

for (jj=0; jj<length; jj++) {
if ( segmentMax < abs(d_vec[jj]) ) segmentMax = abs(d_vec[jj]);
}

return segmentMax;
}
__global__ void segmentMax(float* d_vec, float *segmentMaxes, const int length, const int HighLength, const int HighSegmentLength, const int threadsHigh, const int LowSegmentLength)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
unsigned int startIndex, SegmentLength;

if ( (xIndex*HighSegmentLength > HighLength) & ( (HighLength + (xIndex-threadsHigh+1)*LowSegmentLength) < length ) ){
startIndex = HighLength + (xIndex-threadsHigh)*LowSegmentLength;
SegmentLength = LowSegmentLength;
}
else {
startIndex = xIndex*HighSegmentLength;
SegmentLength = HighSegmentLength;
}
segmentMaxes[xIndex] = getAbsMax(d_vec+startIndex, SegmentLength);
}