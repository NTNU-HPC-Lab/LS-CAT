#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void magnitude(float *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if (xIndex < n) { vec[xIndex] = abs(vec[xIndex]); }
}