#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zero_vector_float(float *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < n )
vec[xIndex]=0.0f;
}