#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void halve_bins(int *bin, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if ( xIndex < n )
bin[xIndex] = bin[xIndex]/2;

}