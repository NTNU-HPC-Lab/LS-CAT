#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  check_support(float * vec_input, float * vec, const int n, int * support_counter)
{
int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if (xIndex < n) {
if ( vec_input[xIndex] != 0 ) {
if (vec[xIndex] != 0) {
atomicAdd(support_counter, 1);
}
}
else {
if (vec[xIndex] == 0) {
atomicAdd(support_counter + 1, 1);
}
}
}
}