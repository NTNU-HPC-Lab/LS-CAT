#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gen_matvecT(float *A, float *x, float *y, const int m, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < n ) {
float c = 0.0f;
for(int i=0; i<m; i++)
c = c + y[i] * A[xIndex * m + i];
x[xIndex] = c;
}
}