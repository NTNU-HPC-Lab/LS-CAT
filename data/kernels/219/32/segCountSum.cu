#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void segCountSum(int *counter, int *segcounter, const int countlength)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if (xIndex < countlength){
for (int jj=0; jj<countlength; jj++){
counter[xIndex] = counter[xIndex] + segcounter[xIndex + jj*countlength];
}
}
}