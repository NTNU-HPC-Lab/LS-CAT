#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_bins(float *vec, int *bin, const int num_bins, const int n, const float slope, const float intercept)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if ( xIndex < n ){
int bin_new_val;
float temp = abs(vec[xIndex]);
if ( temp > (intercept *.000001) ){
bin_new_val=slope * (intercept - temp);
}
else bin_new_val = num_bins;
bin[xIndex]=bin_new_val;
}
}