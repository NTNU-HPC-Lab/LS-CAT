#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zero_vector_int(int *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < n ){
int z=0;
vec[xIndex]=z;
}
}