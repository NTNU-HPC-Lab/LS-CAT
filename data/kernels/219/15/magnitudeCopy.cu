#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void magnitudeCopy(float *mag_vec, float *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

if (xIndex < n) { mag_vec[xIndex] = abs(vec[xIndex]); }
}