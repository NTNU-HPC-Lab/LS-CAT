#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_and_count_bins(float *vec, int *bin, int *bin_counters, const int num_bins, const int n, const float slope, const float intercept)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
float temp = abs(vec[xIndex]);
if ( xIndex < n ){
if ( temp > (intercept *.01) ){
bin[xIndex]=max(0.0f,slope * (intercept - temp));
atomicAdd(bin_counters+bin[xIndex],1);
}
else bin[xIndex] = slope * intercept + 1.0f;
}
}