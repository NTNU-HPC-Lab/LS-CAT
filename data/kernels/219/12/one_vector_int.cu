#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void one_vector_int(int *vec, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < n )
vec[xIndex]=1;
}