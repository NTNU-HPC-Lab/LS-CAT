#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gen_matvec(float *A, float *x, float *y, const int m, const int n)
{
unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
if ( xIndex < m ){
float c = 0.0f;
for(int i=0; i<n; i++)
c = c + x[i] * A[xIndex + m * i];
y[xIndex] = c;
}
}