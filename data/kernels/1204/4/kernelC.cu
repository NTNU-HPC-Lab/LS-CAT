#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelC(int n, float *x, float *y) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride)
if (x[i] > y[i]) {
for (int j = 0; j < n / CONST; j++)
y[i] = x[j] + y[j];
}
}