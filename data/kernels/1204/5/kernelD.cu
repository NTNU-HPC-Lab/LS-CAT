#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelD(int n, float *x, float *y) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
for (int i = index; i < n; i += stride) {
for (int j = 0; j < n / CONST; j++)
y[i] = atomicAdd(&y[j], x[j]);
}
}