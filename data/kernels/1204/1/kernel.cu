#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *array, int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size) {
array[index] += 1.f;
//if (index == 0)
//  printf("### array[%d] = %f\tArray size: %d\n", index, array[index], size);
}
}