#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *array, int size) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size) {
array[index] += 1.f;
if (index == 0)
printf("### Array size: %d\n", size);
}
}