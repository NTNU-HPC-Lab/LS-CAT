#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void k_zero_comp_xyz(float *data, uint n, uint stride)
{
uint i = blockIdx.x * blockDim.x + threadIdx.x;
uint p = blockIdx.y;

if (i < n) {
data[i + p * stride] = 0.f;
}
}