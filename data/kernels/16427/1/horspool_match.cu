#include "hip/hip_runtime.h"
#include "includes.h"
// #pragma once



using namespace std;

#define NUM_THREADS_PER_BLOCK 512

int* create_shifts (char* pattern);

int linear_horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
int num_chunks, int text_size, int pat_len, int myId);


/*
*  Driver function
*  argv[0] is target pattern string
*  argv[1] is text path
*/
__global__ void horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size, int num_chunks, int text_size, int pat_len) {

const int TABLE_SIZ = 126;

int count = 0;
int myId = threadIdx.x + blockDim.x * blockIdx.x;
if(myId > num_chunks){ //if thread is an invalid thread
return;
}

int text_length = (chunk_size * myId) + chunk_size + pat_len - 1;

// don't need to check first pattern_length - 1 characters
int i = (myId*chunk_size) + pat_len - 1;
int k = 0;
while(i < text_length) {
// reset matched character count
k = 0;

if (i >= text_size) {
// break out if i tries to step past text length
break;
}

if (text[i] >= TABLE_SIZ || text[i] < 0) {
// move to next char if unknown char (Unicode, etc.)
++i;
} else {
while(k <= pat_len - 1 && pattern[pat_len - 1 - k] == text[i - k]) {
// increment matched character count
k++;
}
if(k == pat_len) {
// increment pattern count, text index
++count;
++i;

} else {
// add on shift if known char
i = i + shift_table[text[i]];
}
}
}

atomicAdd(num_matches, count);
}