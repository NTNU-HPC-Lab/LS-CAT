#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size, int num_chunks, int text_size, int pat_len , unsigned int* d_output) {
extern __shared__ int s[];


int count = 0;
int myId = threadIdx.x + blockDim.x * blockIdx.x;
if(myId > num_chunks){ //if thread is an invalid thread
return;
}

int text_length = (chunk_size * myId) + chunk_size + pat_len - 1;

// don't need to check first pattern_length - 1 characters
int i = (myId*chunk_size) + pat_len - 1;
int k = 0;
while(i < text_length) {
// reset matched character count
k = 0;

if (i >= text_size) {
// break out if i tries to step past text length
break;
}

while(k <= pat_len - 1 && pattern[pat_len - 1 - k] == text[i - k]) {
// increment matched character count
k++;
}
if(k == pat_len) {
// increment pattern count, text index
++count;
++i;

} else {
i = i + shift_table[text[i]];
}
}

// atomicAdd(num_matches, count);
s[threadIdx.x] = count;
__syncthreads();

// Add count to total matches atomically
if (threadIdx.x == 0 ){
int sum = 0;
for(int idx =0; idx < NUM_THREADS_PER_BLOCK; idx++){
sum += s[idx];
}
d_output[blockIdx.x] = sum;
}
}