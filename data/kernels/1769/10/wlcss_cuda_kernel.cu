#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_2d_cost_matrix){

int32_t params_idx = threadIdx.x;
int32_t template_idx = blockIdx.x;
int32_t stream_idx = blockIdx.y;

int32_t t_len = d_tlen[template_idx];
int32_t s_len = d_slen[stream_idx];

int32_t t_offset = d_toffsets[template_idx];
int32_t s_offset = d_soffsets[stream_idx];

int32_t d_mss_offset = d_mss_offsets[params_idx*gridDim.x*gridDim.y+template_idx*gridDim.y+stream_idx];
int32_t *mss = &d_mss[d_mss_offset];

int32_t *tmp_window = new int32_t[(t_len + 2)]();

int32_t *t = &d_ts[t_offset];
int32_t *s = &d_ss[s_offset];

int32_t reward = d_params[params_idx*3];
int32_t penalty = d_params[params_idx*3+1];
int32_t accepteddist = d_params[params_idx*3+2];

int32_t tmp = 0;

for(int32_t j=0;j<s_len;j++){
for(int32_t i=0;i<t_len;i++){
int32_t distance = d_2d_cost_matrix[s[j]*8 + t[i]];;
if (distance <= accepteddist){
tmp = tmp_window[i]+reward;
} else{
tmp = max(tmp_window[i]-penalty*distance,
max(tmp_window[i+1]-penalty*distance,
tmp_window[t_len+1]-penalty*distance));
}
tmp_window[i] = tmp_window[t_len+1];
tmp_window[t_len+1] = tmp;
}
tmp_window[t_len] = tmp_window[t_len+1];
mss[j] = tmp_window[t_len+1];
tmp_window[t_len+1] = 0;
}
delete [] tmp_window;
}