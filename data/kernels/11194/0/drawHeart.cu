#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void drawHeart(int CIRCLE_SEGMENTS, float *xx, float*yy) {
float scale = 0.5f;

int i = threadIdx.y*CIRCLE_SEGMENTS + threadIdx.x;


float const theta = 2.0f * 3.1415926f * (float)i / (float)CIRCLE_SEGMENTS;
xx[i] = scale * 16.0f * sinf(theta) * sinf(theta) * sinf(theta);
yy[i] = -1 * scale * (13.0f * cosf(theta) - 5.0f * cosf(2.0f * theta) - 2 * cosf(3.0f * theta) - cosf(4.0f * theta));
}