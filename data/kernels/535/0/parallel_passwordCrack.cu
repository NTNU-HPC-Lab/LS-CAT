#include "hip/hip_runtime.h"
#include "includes.h"
/* we need these includes for CUDA's random number stuff */

using namespace std;

#define MAX 26

//int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };
size_t result = 1000 * sizeof(float);

int *a = (int *) malloc(result);

__global__ void parallel_passwordCrack(int length,int*d_output,int *a)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
bool cracked = false;
char alphabetTable[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };
int newB[1000];


__shared__ int nIter;
__shared__ int idT;
__shared__ long totalAttempt;

do{

if(idx == 0){
nIter = 0;
totalAttempt = 0;
}

newB[0]++;
for(int i =0; i<length; i++){
if (newB[i] >= 26 + alphabetTable[i]){
newB[i] -= 26;
newB[i+1]++;
}else break;
}

cracked=true;

for(int k=0; k<length; k++)
{
if(newB[k]!=a[k]){
cracked=false;
break;
}else
{
cracked = true;

}
}
if(cracked && nIter == 0){

idT = idx;
break;
}
else if(nIter){

break;
}

totalAttempt++;
}while(!cracked || !nIter);

if(idx == idT){
for(int i = 0; i< length; i++){

d_output[i] = newB[i];
}

}



}