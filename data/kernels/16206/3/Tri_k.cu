#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Tri_k(float* a, float* b, float* c, float norm, int n)
{
// Identifies the thread working within a group
int tidx = threadIdx.x % n;
// Identifies the data concerned by the computations
int Qt = (threadIdx.x - tidx) / n;
// The global memory access index
int gb_index_x = Qt + blockIdx.x * (blockDim.x / n);

b[gb_index_x * n + tidx] = ((float)tidx + 1.0f) / (norm);
if (tidx > 0 && tidx < n - 1) {
a[gb_index_x * n + tidx] = ((float)tidx + 1.0f) / (norm * 3);
c[gb_index_x * n + tidx] = ((float)tidx + 1.0f) / (norm * 3);
}
else if (tidx == 0) {
a[gb_index_x * n + tidx] = 0.0f;
}
else {
c[gb_index_x * n + tidx] = 0.0f;
}
}