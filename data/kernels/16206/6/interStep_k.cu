#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void interStep_k(float* payoff, size_t scheduleCounter, float dx, float Smin, size_t P1, size_t P2, float barrier)
{
size_t spot_idx = blockIdx.x;
size_t state_idx = threadIdx.x;

// shared memory for the payoff for a fixed spot (each block corresponds to a spot_dx)
//extern __shared__ float shared_payoff_x[];
//shared_payoff_x[state_idx] = payoff[spot_idx + state_idx * gridDim.x];
//__syncthreads();

float temp = 0.0;
size_t P1_k = P1 > scheduleCounter ? P1 - scheduleCounter : 0;
float spot = Smin * expf(spot_idx * dx);

if (state_idx == P2)
{
temp = payoff[spot_idx + P2 * gridDim.x] * (spot >= barrier);
}
else if (P1_k <= state_idx && state_idx < P2)
{
temp = payoff[spot_idx + state_idx * gridDim.x] * (spot >= barrier)
+ payoff[spot_idx + (state_idx + 1) * gridDim.x] * (spot < barrier);
}
else if (state_idx == P1_k)
{
temp = payoff[spot_idx + P1_k * gridDim.x] * (spot < barrier);
}

__syncthreads();
payoff[spot_idx + state_idx * gridDim.x] = temp;
}