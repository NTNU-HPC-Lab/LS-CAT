#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pcr_k(float a, float b, float c, float* y, int n) {
// Identifies the thread working within a group
int tidx = threadIdx.x % n;
// Identifies the data concerned by the computations
int Qt = (threadIdx.x - tidx) / n;
// The global memory access index
int gb_index_x = Qt + blockIdx.x * (blockDim.x / n);
// Local integers
int i, nt, lL, d, tL, tR;
// Local floats
float aL, bL, cL, yL, aLp, bLp, cLp, yLp;
// Shared memory
extern __shared__ float sAds[];

nt = 5 * Qt * n;
d = (n / 2 + (n % 2)) * (tidx % 2) + (int)tidx / 2;
float* sa = (float*)&sAds[nt];
float* sb = (float*)&sa[n];
float* sc = (float*)&sb[n];
float* sy = (float*)&sc[n];
int* sl = (int*)&sy[n];

sa[tidx] = a;
sb[tidx] = b;
sc[tidx] = c;
sy[tidx] = y[gb_index_x * n + tidx];
sl[tidx] = tidx;
__syncthreads();

//Left/Right indices of the reduction
tL = tidx - 1;
if (tL < 0) tL = 0;
tR = tidx + 1;
if (tR >= n) tR = 0;

for (i = 0; i < (int)log2((float)n) + 1; i++) {
lL = (int)sl[tidx];
aL = sa[tidx];
bL = sb[tidx];
cL = sc[tidx];
yL = sy[tidx];

bLp = sb[tL];
//Reduction phase
if (fabsf(aL) > EPS) {
aLp = sa[tL];
cLp = sc[tL];
yLp = sy[tL];

//bL = b[tidx] - a[tidx]*c[tidx]/b[tidx-1];
bL -= aL * cL / bLp;
//yL = y[tidx] - a[tidx]*y[tidx-1]/b[tidx-1];
yL -= aL * yLp / bLp;
//aL = -a[tidx]*a[tidx-1]/b[tidx-1];
aL = -aL * aLp / bLp;

//aL = -aL * aLp / bLp;
}

aLp = sa[tR];
bLp = sb[tR];
cLp = sc[tR];
if (fabsf(aLp) > EPS) {
yLp = sy[tR];
//bL -= c[tidx+1]*a[tidx+1]/b[tidx+1];
bL -= cLp * aLp / bLp;
//yL -= c[tidx+1]*y[tidx+1]/b[tidx+1];
yL -= cLp * yLp / bLp;

cL = -cL * cLp / bLp;
}
__syncthreads();
//Permutation phase
if (i < (int)log2((float)n)) {
sa[d] = aL;
sb[d] = bL;
sc[d] = cL;
sy[d] = yL;
sl[d] = (int)lL;
__syncthreads();
}
}

sy[(int)tidx] = yL / bL;
__syncthreads();
y[gb_index_x * n + sl[tidx]] = sy[tidx];
}