#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tridiag_x_matrix_k(float p_d, float p_m, float p_u, float* u, int n)
{
// Identifies the thread working within a group
int tidx = threadIdx.x % n;
// Identifies the data concerned by the computations
int Qt = (threadIdx.x - tidx) / n;

extern __shared__ float sAds[];
float* su = (float*)&sAds[Qt * n];
su[threadIdx.x] = u[blockIdx.x * blockDim.x + threadIdx.x];
__syncthreads();

float temp;
if (tidx > 0 && tidx < n - 1)
temp = p_d * su[tidx - 1] + p_m * su[tidx] + p_u * su[tidx + 1];
else if (tidx == 0)
temp = p_m * su[tidx] + p_u * su[tidx + 1];
else
temp = p_d * su[tidx - 1] + p_m * su[tidx];

u[blockIdx.x * blockDim.x + threadIdx.x] = temp;
}