#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initPayoff_k(float* payoff, float dx, float Smin, float strike, size_t P1, size_t P2) {
size_t spot_idx = threadIdx.x;
size_t state_idx = blockIdx.x;

float spot = Smin * expf(spot_idx * dx);
size_t idx = spot_idx + state_idx * blockDim.x;

// !! state grid value is equal to state index !!
payoff[idx] = fmaxf(0.0f, spot - strike);// *((state_idx <= P2) && (state_idx >= P1));
}