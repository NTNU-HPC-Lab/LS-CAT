#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void boundaryCondition_k(float* payoff, size_t spotSize, float strike) {
size_t state_idx = threadIdx.x;
payoff[spotSize - 1 + state_idx * spotSize] = 2 * strike;
payoff[0 + state_idx * spotSize] = 0.0;
}