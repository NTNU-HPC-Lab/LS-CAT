#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void polynomial_expansion (float* poly,int degree,int n,float* array)
{
int idx=blockIdx.x*blockDim.x+threadIdx.x;
if(idx<n)
{
float val=0.0;
float exp=1.0;
for(int x=0;x<=degree;++x)
{
val+=exp*poly[x];
exp*=array[idx];
}
array[idx]=val;
}
}