#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
const int igpt =
(gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

if (igpt < n) {
zout[2 * igpt] = din[igpt];
zout[2 * igpt + 1] = 0.0e0;
}
}