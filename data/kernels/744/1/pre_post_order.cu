#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

struct compressed_sparse_column {
int* data;
int* row;
int* column;
int* index_column;
int* index_row_start;
int* index_row_end;
};

struct graph {
compressed_sparse_column* dataset;
bool* roots;
bool* leaves;
bool* singletons;
int vertices;
int edges;
};

__global__ void pre_post_order(int* depth, int* zeta, int* zeta_tilde, graph* dataset_graph) {
int* pre = new int[dataset_graph->vertices];
int* post = new int[dataset_graph->vertices];

memset(pre, 0, dataset_graph->vertices * sizeof(int));
memset(post, 0, dataset_graph->vertices * sizeof(int));

bool* incoming_edges = new bool[dataset_graph->edges];
memset(incoming_edges, false, dataset_graph->edges * sizeof(bool));

bool* q = new bool[dataset_graph->vertices];
memcpy(q, dataset_graph->roots, sizeof(int) * dataset_graph->vertices);

while(true) {
bool* p = new bool[dataset_graph->vertices];
memset(p, false, dataset_graph->vertices * sizeof(bool));
bool global_check = false;

for(int i = 0; i < dataset_graph->vertices; i++) {
if( q[i] ) {
int pre_node = 	pre[i];
int post_node = post[i];

for(int j = dataset_graph->dataset->index_column[i]; dataset_graph->dataset->column[j] == i; j++) {
int neighbor_vertex = dataset_graph->dataset->row[j];
// zeta[i] = undefined!
pre[neighbor_vertex] = pre_node + zeta_tilde[neighbor_vertex];
post[neighbor_vertex] = post_node + zeta_tilde[neighbor_vertex];

incoming_edges[j] = true;
bool flag = true;
for(int k = 0; k < dataset_graph->edges; k++) {
if( dataset_graph->dataset->row[k] == neighbor_vertex && !incoming_edges[k] ) {
flag = false;
break;
}
}
if( flag ) {
global_check = true;
p[neighbor_vertex] = true;
}
}
pre[i] = pre_node + depth[i];
post[i] = post_node + (zeta[i] - 1);
}
}
q = p;
if( !global_check ) {
break;
}
}

}