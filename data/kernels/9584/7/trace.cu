#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void trace(double *diag, int bit) {
unsigned int x = threadIdx.x;
unsigned int mask = 0;

if(bit >= 0) {
mask = 1 << bit;
}

extern __shared__ double s_diag[];
s_diag[x] = diag[x];
__syncthreads();

double a;

for(unsigned int i=1; i < blockDim.x; i <<= 1) {
if(i != mask && i <= x) {
a = s_diag[x-i];

}
__syncthreads();
if(i != mask && i <= x) {
s_diag[x] += a;
}
__syncthreads();
}

__syncthreads();
//copy result back
if(x == 0) {
diag[blockIdx.x] = s_diag[blockDim.x - 1];
return;
}
if(x == 1 && bit >= 0) {
diag[blockIdx.x + 1] = s_diag[blockDim.x - 1 - mask];
return;
}
}