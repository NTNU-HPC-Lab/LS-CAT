#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void two_qubit_ptm(double *dm, double *ptm_g, unsigned int bit0, unsigned int bit1, unsigned int no_qubits) {
const unsigned int x = threadIdx.x;
const unsigned int high_x = blockIdx.x * blockDim.x;



extern __shared__ double ptm[];
double *data = &ptm[256]; //need blockDim.x double floats

// the lowest to bits of x are used to address bit0, the next two are used to address bit1
// global address = <- pos =
// aaaxxbbbbyycccc  <- aaabbbbccccxxyy

int higher_bit = max(bit0, bit1);
int lower_bit = min(bit0, bit1);
int high_mask = ~ ( (1 << (2*higher_bit+2)) - 1 ); //a mask (of pos)
int mid_mask = (~ ( (1 << (2*lower_bit + 4)) - 1)) & (~high_mask);  //b mask
int low_mask  = ~(high_mask | mid_mask) & (~0xf);  //c mask

int pos = high_x | x;
int global_from =
(pos & high_mask)
| ((pos & mid_mask) >> 2)
| ((pos & low_mask) >> 4)
| ((pos & 0x3) << (2 * bit0))
| (((pos & 0xc) >>2)  << (2 * bit1));

//fetch ptm to shared memmory
//need to fetch several values per thread if blockDim.x is less than 256 (only for small dms...)
for(int i=0; i < 256; i+=blockDim.x) {
if(i+x < 256) {
ptm[i+x] = ptm_g[i+x];
}
}
if (high_x + x >= (1 << (2*no_qubits))) return;


//fetch data block to shared memory
data[x] = dm[global_from];
__syncthreads();

unsigned int row = x & 0xf;
unsigned int idx = x & ~0xf;

double acc=0;
for(int i=0; i<16; i++) {
acc += ptm[16*row + i]*data[idx+i];
}


__syncthreads();
dm[global_from] = acc;

}