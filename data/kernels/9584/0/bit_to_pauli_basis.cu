#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void bit_to_pauli_basis(double *complex_dm, unsigned int mask, unsigned int no_qubits) {
const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

const double sqrt2 =  0.70710678118654752440;
//const double sqrt2 =  1;

if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;

int b_addr = ((x|mask)<<no_qubits | (y&~mask)) << 1;
int c_addr = ((x&~mask)<<no_qubits | (y|mask)) << 1;

if (x&mask && (~y&mask)){
double b = complex_dm[b_addr];
double c = complex_dm[c_addr];
complex_dm[b_addr] = (b+c)*sqrt2;
complex_dm[c_addr] = (b-c)*sqrt2;
}
if ((~x&mask) && (y&mask)){
b_addr+=1;
c_addr+=1;
double b = complex_dm[b_addr];
double c = complex_dm[c_addr];
complex_dm[b_addr] = (b+c)*sqrt2;
complex_dm[c_addr] = (b-c)*sqrt2;
}
}