#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void dm_reduce(double *dm, unsigned int bit, double *dm0, unsigned int state, unsigned int no_qubits) {

const int addr = blockIdx.x*blockDim.x + threadIdx.x;

if(addr >= (1<< (2*no_qubits))) return;

const int low_mask = (1 << (2*bit))-1;      //0000011111
const int high_mask = (~low_mask) << 2;     //1110000000

if(((addr >> (2*bit)) & 0x3) == state*0x3) {
dm0[ (addr & low_mask) | ((addr & high_mask) >> 2) ] = dm[addr];
}
}