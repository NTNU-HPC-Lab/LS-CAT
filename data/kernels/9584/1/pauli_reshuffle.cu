#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void pauli_reshuffle(double *complex_dm, double *real_dm, unsigned int no_qubits, unsigned int direction) {

const int x = (blockIdx.x *blockDim.x) + threadIdx.x;
const int y = (blockIdx.y *blockDim.y) + threadIdx.y;

if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;


//do we need imaginary part? That is the case if we have an odd number of bits for y in our adress (bit in y is 1, bit in x is 0)
unsigned int v = ~x & y;


unsigned int py = 0;
while (v) {
py += v&1;
v >>= 1;
}

py = py & 0x3;

//short version: while (v>1) { v = (v >> 1) ^ v ;}
//bit bang version
/*v ^= v >> 1;*/
/*v ^= v >> 2;*/
/*v = (v & 0x11111111U) * 0x11111111U;*/
/*v = (v >> 28) & 1;*/

const unsigned int addr_complex = (((x << no_qubits) | y) << 1) + (py&1);


//the adress in pauli basis is obtained by interleaving
unsigned int addr_real = 0;
for (int i = 0; i < 16; i++) {
addr_real |= (x & 1U << i) << i | (y & 1U << i) << (i + 1);
}


if(direction == 0) {
real_dm[addr_real] = ((py==3 || py==2)? -1 : 1)*complex_dm[addr_complex];
}
else {
complex_dm[addr_complex] = ((py==3 || py == 2)? -1 : 1)*real_dm[addr_real];
}
}