#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void swap(double *dm, unsigned int bit1, unsigned int bit2, unsigned int no_qubits) {
unsigned int addr = threadIdx.x + blockDim.x*blockIdx.x;

if (addr >= (1<<2*no_qubits)) return;

unsigned int bit1_mask = (0x3 << (2*bit1));
unsigned int bit2_mask = (0x3 << (2*bit2));

unsigned int addr2 = ( addr & ~(bit1_mask | bit2_mask)) |
((addr & bit1_mask) << (2*(bit2 - bit1))) |
((addr & bit2_mask) >> (2*(bit2 - bit1)));

double t;
if (addr > addr2) {
t = dm[addr2];
dm[addr2] = dm[addr];
dm[addr] = t;
}
}