#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void single_qubit_ptm(double *dm, double *ptm_g,  unsigned int bit, unsigned int no_qubits) {
const unsigned int x = threadIdx.x;
const unsigned int high_x = blockIdx.x * blockDim.x;

if (high_x + x >= (1 << (2*no_qubits))) return;

//the two lowest bits of thread id are used to index the target bit,
//      xx <- target bit
int high_mask = ~ ( (1 << (2*bit+2)) - 1 ); // 1111100000000
int low_mask  = ~high_mask & (~0x3);        // 0000011111100

int pos = high_x | x;
int global_from = (pos & high_mask) | ((pos & 0x3) << (2*bit)) | ((pos & low_mask)>>2);

extern __shared__ double ptm[];
double *data = &ptm[16]; //need blockDim.x double floats

//first fetch the transfer matrix to shared memory
if(x < 16) ptm[x] = ptm_g[x];

if(no_qubits < 2) { //what a boring situation
ptm[x+4] = ptm_g[x+4];
ptm[x+8] = ptm_g[x+8];
ptm[x+12] = ptm_g[x+12];
}

//fetch block to shared memory
data[x] = dm[global_from];
__syncthreads();

//do calculation

int row = x & 0x3;
int idx = x & ~0x3;

double acc = 0;

acc += ptm[4*row    ] * data[idx    ];
acc += ptm[4*row + 1] * data[idx + 1];
acc += ptm[4*row + 2] * data[idx + 2];
acc += ptm[4*row + 3] * data[idx + 3];

//upload back to global memory
__syncthreads();
dm[global_from] = acc;
}