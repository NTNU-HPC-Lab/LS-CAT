#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void get_diag(double *dm9, double *out, unsigned int no_qubits) {
int x = (blockIdx.x *blockDim.x) + threadIdx.x;

if (x >= (1 << no_qubits)) return;
unsigned int addr_real = 0;
for (int i = 0; i < 16; i++) {
addr_real |= (x & 1U << i) << i | (x & 1U << i) << (i + 1);
}
out[x] = dm9[addr_real];
}