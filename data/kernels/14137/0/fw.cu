#include "hip/hip_runtime.h"
#include "includes.h"
/*
* /usr/local/cuda/bin/nvcc -gencode arch=compute_20,code=compute_20 -o fw_kernel.ptx -ptx fw_kernel.cu
*/

extern "C" {


}
__global__ void fw(float *adj_array, int *next_array, int k, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockDim.y * blockIdx.y + threadIdx.y;

float check;
float next;
if (i < N && j < N)
{
check = adj_array[j * N + k] + adj_array[k * N + i];
next = next_array[j * N + k];
}

__syncthreads();
if (i == 0 || j == 0 || i > N || j > N) return;

if (check < adj_array[j * N + i])
{
adj_array[j * N + i] = check;
next_array[j * N + i] = next;
}
}