#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_array_gpu(float *a, int power, int array_size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

float result=1;
if (idx<array_size)
{
for(int i=0; i<power; ++i)
result*=a[idx];
a[idx] = result;
}
}