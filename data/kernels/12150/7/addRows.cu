#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addRows(double *matrix, int *d_i){
int i=*d_i;
int n=blockDim.x+i;
int id= n*(blockIdx.x+i+1) + threadIdx.x+i;
__shared__ double multiplier;

if(threadIdx.x==0){
multiplier=matrix[n*(blockIdx.x+1+i)+i]/matrix[n*i+i];
}
__syncthreads();

matrix[id]-=matrix[n*i+threadIdx.x+i]*multiplier;
}