#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addPermutations(double *determinant, double *permutations, int *n){

int nn=*n**n-1;
*determinant=0;
for(int i=0;i<nn;i++){
*determinant+=permutations[i];
}
}