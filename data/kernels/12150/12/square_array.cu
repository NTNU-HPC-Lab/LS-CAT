#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int array_size)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx<array_size) a[idx] = a[idx] * a[idx];
}