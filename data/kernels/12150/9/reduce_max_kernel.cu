#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_max_kernel(float *d_out, const float *d_logLum, int size) {

int tid         = threadIdx.x;                              // Local thread index
int myId        = blockIdx.x * blockDim.x + threadIdx.x;    // Global thread index

extern __shared__ float temp[];

// --- Loading data to shared memory. All the threads contribute to loading the data to shared memory.
temp[tid] = (myId < size) ? d_logLum[myId] : -10000000;

// --- Your solution
// if (myId < size) { temp[tid] = d_logLum[myId]; } else { temp[tid] = d_logLum[tid]; }

// --- Before going further, we have to make sure that all the shared memory loads have been completed
__syncthreads();

// --- Reduction in shared memory. Only half of the threads contribute to reduction.
for (unsigned int s=blockDim.x/2; s>0; s>>=1)
{
if (tid < s) { temp[tid] = fmaxf(temp[tid], temp[tid + s]); }
// --- At the end of each iteration loop, we have to make sure that all memory operations have been completed
__syncthreads();
}

// --- Your solution
//for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
//    if (tid < s) { if (myId < size) { temp[tid] = fmaxf(d_logLum[myId + s], d_logLum[myId]); } else { temp[tid] = d_logLum[tid]; } }
//    __syncthreads();
//}

if (tid == 0) {
d_out[blockIdx.x] = temp[0];
}
}