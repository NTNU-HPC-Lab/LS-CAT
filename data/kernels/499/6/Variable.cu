#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include "../ErrorChecking/ErrorChecking.cu"

///////////////////////////////////////////////////////////////////////
////////////////////////HOST SIDE//////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct HostVariable{
	
	int* dMem;						//ptr to memory
	int domainSize;					//variable size (cardinality)

	__host__ HostVariable(int); 	//allocate memory
	__host__ int* getPtr();			//return memory ptr;
	__host__ ~HostVariable();		//deallocate
};

///////////////////////////////////////////////////////////////////////

__host__ HostVariable::HostVariable(int dm):
	domainSize(dm){
	ErrorChecking::hostMessage("Warn::HostVariable::HostVariable::ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&dMem,sizeof(int)*domainSize),"HostVariable::HostVariable");
}

///////////////////////////////////////////////////////////////////////

__host__ HostVariable::~HostVariable(){
	ErrorChecking::hostMessage("Warn::HostVariable::~HostVariable::DEALLOCATION");
	hipFree(dMem);
}

///////////////////////////////////////////////////////////////////////

__host__ int* HostVariable::getPtr()
	{return dMem;}


///////////////////////////////////////////////////////////////////////
////////////////////////DEVICE SIDE////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct DeviceVariable{
	int ground;			//track if variable is ground
	int changed;		//track if variable was modified
	int failed;			//track if variable is in a failed state
	int domainSize;		//size of the domain

	int* domain;		//ptr to domain memory

	int fullParallel;	//choose always parallel code execution 

	__device__ DeviceVariable();			//do nothing
	__device__ DeviceVariable(int*,int); 	//initialize
	__device__ void init(int*, int);		//initialize
	__device__ void init2(int*, int);		//initialize without setting
											//assume already setted memory
	__device__ ~DeviceVariable();			//do nothing

	__device__ int assign(int);			//assign choesen variable and returns 0.
										//otherwise -1
	__device__ int undoAssign(int);		//undo assignement
	__device__ void addTo(int,int);		//increment or decrement by delta

	__device__ void checkGround();		//check if variable is in ground state and modify ground
	__device__ void checkFailed();		//check if variable is in failed state and modify failed

	__device__ void print();			//stampa with modes

};

///////////////////////////////////////////////////////////////////////

__device__ inline DeviceVariable::DeviceVariable(){}

///////////////////////////////////////////////////////////////////////

__device__ inline DeviceVariable::~DeviceVariable(){}

///////////////////////////////////////////////////////////////////////

__device__ inline DeviceVariable::DeviceVariable(int* dMem, int ds):
	domainSize(ds),ground(-1),changed(-1),failed(-1),fullParallel(true),
	domain(dMem){
		for(int i = 0; i < domainSize; ++i)dMem[i]=1;
	}

///////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::init(int* dMem, int ds){
	domainSize = ds;
	domain = dMem;
	fullParallel = true;
	ground  = -1;
	changed = -1;
	failed  = -1;

	for(int i = 0; i < domainSize; ++i)dMem[i]=1;
}

///////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::init2(int* dMem, int ds){

	domainSize = ds;
	domain = dMem;
	fullParallel = true;
	ground  = -1;
	changed = -1;
	failed  = -1;
}

///////////////////////////////////////////////////////////////////////


__device__ inline void externAssignSequential(int* domain, int size, int value){

	for(int i = 0; i < size; ++i){
		if(i != value)--domain[i];
	}

}

__device__ void externAssignParallel(int* domain, int size, int value){

	if(threadIdx.x + blockIdx.x * blockDim.x < size && 
	   threadIdx.x + blockIdx.x * blockDim.x != value)
		--domain[threadIdx.x + blockIdx.x * blockDim.x];

}

__device__ inline int DeviceVariable::assign(int value){

/*	if(value < 0 || value >= domainSize){
		ErrorChecking::deviceError("Error::Variable::assign::ASSIGNMENT OUT OF BOUND");
		return -1;
	}

	if(failed == 1){
		ErrorChecking::deviceError("Error::Variable::assign::VARIABLE ALREADY FAILED");
		return -1;
	}

	if(domain[value]<=0){
		ErrorChecking::deviceError("Error::Variable::assign::VALUE NO MORE IN DOMAIN");
		return -1;
	}

	if(ground >= 0 && value != ground){
		ErrorChecking::deviceError("Error::Variable::assign::VARIABLE NOT GROUND");
		return -1;
	}*/


	externAssignParallel(domain, domainSize, value);

	ground = value;
	return 0;

}		

///////////////////////////////////////////////////////////////////////

__device__ inline void externUndoAssignSequential(int* domain, int size, int value){

	for(int i = 0; i < size; ++i){
		if(i != value)++domain[i];
	}

}

__global__ void externUndoAssignParallel(int* domain, int size, int value){

	if(threadIdx.x + blockIdx.x * blockDim.x < size && 
	   threadIdx.x + blockIdx.x * blockDim.x != value)
		++domain[threadIdx.x + blockIdx.x * blockDim.x];

}

__device__ inline int DeviceVariable::undoAssign(int value){

/*	if(value < 0 || value >= domainSize){
		ErrorChecking::deviceError("Error::Variable::undoAssign::OUT OF BOUND");
		return -1;
	}

	if(ground == -1){
		ErrorChecking::deviceError("Error::Variable::undoAssign::VARIABLE NOT GROUND");
		return -1;
	}*/

	hipStream_t s;
	ErrorChecking::deviceErrorCheck(hipStreamCreateWithFlags(&s, hipStreamNonBlocking),"DeviceVariable::undoAssign");
	externUndoAssignParallel<<<1,domainSize>>>(domain, domainSize, value);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceVariable::undoAssign");
	ErrorChecking::deviceErrorCheck(hipStreamDestroy(s),"DeviceVariable::undoAssign");
	ErrorChecking::deviceErrorCheck(hipDeviceSynchronize(),"DeviceVariable::undoAssign");

	checkGround();

	return 0;

}

///////////////////////////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::addTo(int value, int delta){
/*	if(value < 0 || value >= domainSize){

		ErrorChecking::deviceError("Error::Variable::addTo::ADDING OUT OF BOUND");
		return;
	}*/
	
	if(domain[value] > 0 && domain[value] + delta <= 0) changed = 1;

	domain[value]+=delta;

	checkGround();
	checkFailed();
	
}

///////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::checkGround(){

	int sum = 0;
	for(int i = 0; i < domainSize; ++i){
		if(domain[i]==1){
			++sum;
			ground = i;
		}
	}
	if(sum != 1) ground = -1;

}

///////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::checkFailed(){

	for(int i = 0; i < domainSize; ++i)
		if(domain[i]==1){
			failed = -1;
			return;
		}
	failed = 1;

}

///////////////////////////////////////////////////////////////////////

__device__ inline void DeviceVariable::print(){

	for (int i = 0; i < domainSize; ++i){
		if(domain[i] == 0)
			printf("\033[31m%d\033[0m ", domain[i]);
		else if(domain[i] > 0)printf("\033[34m%d\033[0m ", domain[i]);
		else if(domain[i] < 0)printf("\033[31m%d\033[0m ", -domain[i]);
	}

	if(ground >= 0)printf(" ::: \033[32mgrd:%d\033[0m ", ground);
	else printf(" ::: grd:%d ", ground);

	if(changed == 1)printf("\033[31mchd:%d\033[0m ", changed);
	else printf("chd:%d ", changed);

	if(failed == 1)printf("\033[31mfld:%d\033[0m ", failed);
	else printf("fld:%d ", failed);

	printf("sz:%d ", domainSize);

	printf("ptr:%d\n", domain);
}

///////////////////////////////////////////////////////////////////////
