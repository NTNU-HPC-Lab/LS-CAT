#include "hip/hip_runtime.h"
#pragma once
#include "../VariableCollection/VariableCollection.cu"
#include "../ErrorChecking/ErrorChecking.cu"

struct DeviceQueenPropagation{

	//////////////////////////////////////SINGLE THREAD//////////////////////////////////////

	__device__ int static inline nextAssign(DeviceVariableCollection&,int);		//assign next value not already tried
																	//returns assigned value

	__device__ int static inline allDifferent(DeviceVariableCollection&,int,int,int);		//propagate for all different constraint code 3
	__device__ int static inline diagDifferent(DeviceVariableCollection&,int,int,int);	//propagate for diag constraint code 4

	__device__ int static inline sequentialForwardChecking(DeviceVariableCollection&,int,int);	//csp forward propagation code 5
	__device__ int static inline sequentialBacktracking(DeviceVariableCollection&);		//csp undo forward propagation

	//////////////////////////////////////MULTI THREAD//////////////////////////////////////

	__device__ int static inline parallelForwardChecking(DeviceVariableCollection&,int,int);
	__device__ int static inline parallelForwardChecking(DeviceVariableCollection&,int,int,hipStream_t&);
	__device__ int static inline parallelBacktracking(DeviceVariableCollection&);		//csp undo forward propagation
};

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::nextAssign(DeviceVariableCollection& vc, int var){

/*	if(var < 0 || var >= vc.nQueen){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::nextAssign::VAR OUT OF BOUND");
		return -1;
	}

	if(vc.lastValues[var] >= vc.nQueen){
		if(threadIdx.x == 0)ErrorChecking::deviceMessage("Warn::DeviceQueenPropagation::nextAssign::VALUE OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].failed == 1){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::nextAssign::VAR ALREADY FAILED");
		return -1;
	}*/

	__shared__ int nextAss;
	nextAss = -1;

	__syncthreads();

	if(threadIdx.x == 0){
		int next;
		for(next = vc.lastValues[var];next<vc.nQueen;++next){
			if(vc.deviceVariable[var].domain[next]==1){
				vc.lastValues[var]=next+1;
				nextAss = next;
				break;
			}
		}
	}

	__syncthreads();

	if(nextAss != -1){

		if(threadIdx.x < vc.nQueen && threadIdx.x != nextAss){
			--vc.deviceVariable[var].domain[threadIdx.x];
		}

		if(threadIdx.x == 0){
			vc.deviceVariable[var].ground = nextAss;
		}

	}

	//if(threadIdx.x == 0)ErrorChecking::deviceMessage("Warn::DeviceQueenPropagation::nextAssign::NEXTVALUE NOT FOUND");


	return nextAss;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::allDifferent(DeviceVariableCollection& vc, int var, int val, int delta){

/*	if(var < 0 || var > vc.nQueen || val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::allDifferent::OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::QueenPropagation::allDifferent::VARIABLE NOT GROUND");
		return -1;
	}*/
	
	for(int i = 0; i < vc.nQueen; ++i)
		if(i != var){
			vc.deviceVariable[i].addTo(val,delta);

		}
	
	if(delta < 0)vc.deviceQueue.add(var,val,3);

	return 0;	

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::diagDifferent(DeviceVariableCollection& vc, int var, int val, int delta){

/*	if(var < 0 || var > vc.nQueen || val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::diagDifferent::OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::diagDifferent::VARIABLE NOT GROUND");
		return -1;
	}*/

	int i=var+1,j=val+1;
	while(i<vc.nQueen && j<vc.nQueen){
		vc.deviceVariable[i].addTo(j,delta);
		++i;++j;
	}

	i=var-1,j=val-1;
	while(i>=0 && j>=0){
		vc.deviceVariable[i].addTo(j,delta);
		--i;--j;
	}

	i=var-1,j=val+1;
	while(i>=0 && j<vc.nQueen){
		vc.deviceVariable[i].addTo(j,delta);
		--i;++j;
	}

	i=var+1,j=val-1;
	while(i<vc.nQueen && j>=0){
		vc.deviceVariable[i].addTo(j,delta);
		++i;--j;
	}

	if(delta < 0)vc.deviceQueue.add(var,val,4);
	return 0;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::sequentialForwardChecking(DeviceVariableCollection& vc, int var, int val){

/*	if(var < 0 || var > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::sequentialForwardChecking:: VAR OUT OF BOUND");
		return -1;
	}

	if(val < 0 || val > vc.nQueen){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::sequentialForwardChecking:: VAL OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::sequentialForwardChecking::VARIABLE NOT GROUND");
		return -1;
	}*/

	allDifferent(vc,var,val,-1);
	diagDifferent(vc,var,val,-1);

	bool ch = false;
	do{
		ch=false;
		for(int i = 0; i < vc.nQueen; ++i){
			if(vc.deviceVariable[i].changed==1){
				if(vc.deviceVariable[i].ground>=0){
					allDifferent(vc,i,vc.deviceVariable[i].ground,-1);
					diagDifferent(vc,i,vc.deviceVariable[i].ground,-1);
					ch = true;
				}
				vc.deviceVariable[i].changed=-1;
			}
		}
	}while(ch);

	vc.deviceQueue.add(var,val,5);

	if (vc.isFailed()) return -1;

	return 0;

}

////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::sequentialBacktracking(DeviceVariableCollection& vc){

/*	if(vc.deviceQueue.front()->cs!=5){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::sequentialBacktracking::ERROR IN QUEUE");
		return -1;		
	}

	if(vc.deviceQueue.empty()){
		ErrorChecking::deviceError("Error::DeviceQueenPropagation::sequentialBacktracking::EMPTY QUEUE");
		return -1;		
	}*/

	int t1=vc.deviceQueue.front()->var;
	int t2=vc.deviceQueue.front()->val;

	for(int i = t1+1; i < vc.nQueen; ++i)vc.lastValues[i]=0;

	vc.deviceQueue.pop();
	while(vc.deviceQueue.front()->cs!=5){
		switch(vc.deviceQueue.front()->cs){
			case 3:{
				allDifferent(vc,vc.deviceQueue.front()->var,vc.deviceQueue.front()->val,+1);	
			}break;
			case 4:{
				diagDifferent(vc,vc.deviceQueue.front()->var,vc.deviceQueue.front()->val,+1);	
			}break;
		}
		vc.deviceQueue.pop();

		if(vc.deviceQueue.empty())break;
	}

	vc.deviceVariable[t1].undoAssign(t2);
	return 0;

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceQueenPropagation::parallelForwardChecking(DeviceVariableCollection& vc, int var, int val){

/*	if(var < 0 || var > vc.nQueen){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VAR OUT OF BOUND");
		return -1;
	}

	if(val < 0 || val > vc.nQueen){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VAL OUT OF BOUND");
		return -1;
	}

	if(vc.deviceVariable[var].ground != val){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelForwardPropagation::VARIABLE NOT GROUND");
		return -1;
	}

	__syncthreads();*/

	{
		int columnIndex = threadIdx.x % vc.nQueen;
		int rowIndex = int(threadIdx.x/vc.nQueen);

		__shared__ bool ch;

		if(threadIdx.x < vc.nQueen*vc.nQueen){
			if(rowIndex != var && val == columnIndex){

				int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex],-1);
				if(old == 1){
					vc.deviceVariable[rowIndex].changed = 1;
				}

			}
			
			if(rowIndex != var && columnIndex == rowIndex && columnIndex+val-var < vc.nQueen && columnIndex+val-var >= 0){

				int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex+val-var],-1);
				if(old == 1){
					vc.deviceVariable[rowIndex].changed = 1;
				}

			}
			
			if(rowIndex != var && vc.nQueen-columnIndex == rowIndex && columnIndex-(vc.nQueen-val)+var < vc.nQueen && columnIndex-(vc.nQueen-val)+var >= 0){

				int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex-(vc.nQueen-val)+var],-1);
				if(old == 1){
					vc.deviceVariable[rowIndex].changed = 1;
				}

			}
		}
		__syncthreads();

		if(threadIdx.x == 0){
			int old = atomicAdd(&vc.deviceQueue.count,1);
			vc.deviceQueue.q[old].var = var;
			vc.deviceQueue.q[old].val = val;
			vc.deviceQueue.q[old].cs = 6;

		}

		if(threadIdx.x >= vc.nQueen && threadIdx.x < vc.nQueen*2)
			vc.deviceVariable[threadIdx.x-vc.nQueen].checkFailed();

		if(threadIdx.x >= vc.nQueen*2 && threadIdx.x < vc.nQueen*3)
			vc.deviceVariable[threadIdx.x-vc.nQueen*2].checkGround();

		do{
			
			__syncthreads();
			
			ch=false;
			
			for(int i = var+1; i < vc.nQueen; ++i){


				if(vc.deviceVariable[i].changed == 1){

					if(vc.deviceVariable[i].ground>=0){

						__syncthreads();

						if(threadIdx.x < vc.nQueen*vc.nQueen){
							if(rowIndex != i && vc.deviceVariable[i].ground == columnIndex){

								int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex],-1);
								if(old == 1){
									vc.deviceVariable[rowIndex].changed = 1;
								}

							}
							
							if(rowIndex != i && columnIndex == rowIndex && columnIndex+vc.deviceVariable[i].ground-i < vc.nQueen && columnIndex+vc.deviceVariable[i].ground-i >= 0){

								int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex+vc.deviceVariable[i].ground-i],-1);
								if(old == 1){
									vc.deviceVariable[rowIndex].changed = 1;
								}

							}
							
							if(rowIndex != i && vc.nQueen-columnIndex == rowIndex && columnIndex-(vc.nQueen-vc.deviceVariable[i].ground)+i < vc.nQueen && columnIndex-(vc.nQueen-vc.deviceVariable[i].ground)+i >= 0){

								int old = atomicAdd(&vc.deviceVariable[rowIndex].domain[columnIndex-(vc.nQueen-vc.deviceVariable[i].ground)+i],-1);
								if(old == 1){
									vc.deviceVariable[rowIndex].changed = 1;
								}

							}
						}

						__syncthreads();

						if(threadIdx.x == 0){
							int old = atomicAdd(&vc.deviceQueue.count,1);
							vc.deviceQueue.q[old].var = i;
							vc.deviceQueue.q[old].val = vc.deviceVariable[i].ground;
							vc.deviceQueue.q[old].cs = 6;

						}

						if(threadIdx.x >= vc.nQueen && threadIdx.x < vc.nQueen*2)
							vc.deviceVariable[threadIdx.x-vc.nQueen].checkFailed();

						if(threadIdx.x >= vc.nQueen*2 && threadIdx.x < vc.nQueen*3)
							vc.deviceVariable[threadIdx.x-vc.nQueen*2].checkGround();

						ch = true;
					}

					__syncthreads();
					vc.deviceVariable[i].changed=-1;
				}
			}

			if(vc.isFailed())ch = false;

		}while(ch);

	}

	__syncthreads();

	if(vc.isFailed()){
		__syncthreads();
		if(threadIdx.x < vc.nQueen)vc.deviceVariable[threadIdx.x].changed = -1;
		if(threadIdx.x == 0)vc.deviceQueue.add(var,val,5);
		__syncthreads();
		return 1;
	}

	__syncthreads();
	if(threadIdx.x == 0)vc.deviceQueue.add(var,val,5);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int inline DeviceQueenPropagation::parallelBacktracking(DeviceVariableCollection& vc){

/*	if(vc.deviceQueue.front()->cs!=5){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelBacktracking::ERROR IN QUEUE");
		return -1;		
	}

	if(vc.deviceQueue.empty()){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceQueenPropagation::parallelBacktracking::EMPTY QUEUE");
		return -1;		
	}*/

	__shared__ int t1;
	__shared__ int t2;

	t1 = vc.deviceQueue.front()->var;
	t2 = vc.deviceQueue.front()->val;

	__syncthreads();

	if(threadIdx.x == 0){
		vc.deviceQueue.pop();
	}

	if(threadIdx.x >= t1+1 && threadIdx.x < vc.nQueen){
		vc.lastValues[threadIdx.x]=0;
	}

	__syncthreads();

	while(vc.deviceQueue.front()->cs!=5 && !vc.deviceQueue.empty()){

		int col = threadIdx.x % vc.nQueen;
		int row = int(threadIdx.x/vc.nQueen);

		int var = vc.deviceQueue.front()->var;
		int val = vc.deviceQueue.front()->val;

		if(threadIdx.x < vc.nQueen*vc.nQueen){
			if(row != var && val == col){
				atomicAdd(&vc.deviceVariable[row].domain[col],1);
			}
			
			if(row != var && col == row && col+val-var < vc.nQueen && col+val-var >= 0){
				atomicAdd(&vc.deviceVariable[row].domain[col+val-var],1);
			}
			
			if(row != var && vc.nQueen-col == row && col-(vc.nQueen-val)+var < vc.nQueen && col-(vc.nQueen-val)+var >= 0){
				atomicAdd(&vc.deviceVariable[row].domain[col-(vc.nQueen-val)+var],1);
			}
		}

		__syncthreads();

		if(threadIdx.x == 0)vc.deviceQueue.pop();

		__syncthreads();
	}

	if(threadIdx.x < vc.nQueen && threadIdx.x != t2){
		++vc.deviceVariable[t1].domain[threadIdx.x];
	}

	__syncthreads();

	if(threadIdx.x < vc.nQueen) vc.deviceVariable[threadIdx.x].checkFailed();
	if(threadIdx.x >= vc.nQueen && threadIdx.x <2*vc.nQueen) vc.deviceVariable[threadIdx.x-vc.nQueen].checkGround();

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
