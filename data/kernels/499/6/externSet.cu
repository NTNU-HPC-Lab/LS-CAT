#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void externSet(int* variablesMem,int* lastValuesMem, int nQueen,int nVariableCollection){

int index = threadIdx.x + blockIdx.x * blockDim.x;
if(index < nVariableCollection*nQueen*nQueen){
variablesMem[index] = 1;
if(index < nVariableCollection*nQueen)
lastValuesMem[index] = 0;
}

}