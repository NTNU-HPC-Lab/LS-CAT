#include "hip/hip_runtime.h"
#pragma once
#include "../Variable/Variable.cu"
#include "../TripleQueue/TripleQueue.cu"
#include "../VariableCollection/VariableCollection.cu"
#include "../QueenPropagation/QueenPropagation.cu"
#include "../ErrorChecking/ErrorChecking.cu"
#include <cstdio>

///////////////////////////////////////////////////////////////////////
////////////////////////HOST SIDE//////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct HostParallelQueue{

	DeviceVariableCollection* deviceVariableCollection;

	DeviceVariable* deviceVariable;
	int* variablesMem;
	int* lastValuesMem;
	int* lockReading;
	int* levelLeaved;
	Triple* tripleQueueMem;

	int size;
	int nQueen;

	__host__ HostParallelQueue(int,int);
	__host__ ~HostParallelQueue();
};

//////////////////////////////////////////////////////////////////////////////////////////////

__host__ HostParallelQueue::HostParallelQueue(int nq, int sz):nQueen(nq),size(sz){
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&lockReading,sizeof(int)*size),"Error::HostParallelQueue::ALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&deviceVariableCollection,sizeof(DeviceVariableCollection)*size),"Error::HostParallelQueue::ALLOCATE 2");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&levelLeaved,sizeof(int)*size),"Error::HostParallelQueue::ALLOCATE 3");

	ErrorChecking::hostErrorCheck(hipMalloc((void**)&deviceVariable,sizeof(DeviceVariable)*size*nQueen),"HostParallelQueue::DEVICE VARIABLE ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&variablesMem,sizeof(int)*nQueen*nQueen*size),"HostParallelQueue::VARIABLE MEM ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&lastValuesMem,sizeof(int)*nQueen*size),"HostParallelQueue::LAST VALUES MEM ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&tripleQueueMem,sizeof(Triple)*nQueen*nQueen*3*size),"HostParallelQueue::TRIPLE QUEUE MEM ALLOCATION");

	ErrorChecking::hostErrorCheck(hipPeekAtLastError(),"HostParallelQueue::EXTERN SET CALL");
	ErrorChecking::hostErrorCheck(hipDeviceSynchronize(),"HostParallelQueue::SYNCH");

}

//////////////////////////////////////////////////////////////////////////////////////////////

__host__ HostParallelQueue::~HostParallelQueue(){
	ErrorChecking::hostErrorCheck(hipFree(levelLeaved),"Error::hostParallelQueue::DEALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipFree(lockReading),"Error::hostParallelQueue::DEALLOCATE 1");
	ErrorChecking::hostErrorCheck(hipFree(deviceVariableCollection),"Error::hostParallelQueue::DEALLOCATE 2");
	ErrorChecking::hostErrorCheck(hipFree(variablesMem),"Error::hostParallelQueue::VARIABLES MEM DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(lastValuesMem),"Error::hostParallelQueue::LAST VALUES MEM DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(tripleQueueMem),"Error::hostParallelQueue::TRIPLE QUEUE ME DEALLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(deviceVariable),"Error::hostParallelQueue::DEVICE VARIABLE DEALLOCATION");
}

///////////////////////////////////////////////////////////////////////
////////////////////////DEVICE SIDE////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct DeviceParallelQueue{
	int size;									//max number of element(fixed)
	int nQueen;									//size of csp
	int maxUsed;
	int lockMaxUsed;

	DeviceVariableCollection* deviceVariableCollection;
	DeviceVariable* deviceVariable;
	int* lockReading;
	int* variablesMem;
	int* lastValuesMem;
	int* levelLeaved;
	Triple* tripleQueueMem;

	__device__ DeviceParallelQueue();					//do nothing
	__device__ DeviceParallelQueue(DeviceVariableCollection*,DeviceVariable*,int*,int*,int*,int*,Triple*,int,int);	//initialize
	__device__ void init(DeviceVariableCollection*,DeviceVariable*,int*,int*,int*,int*,Triple*,int,int);			//initialize

	__device__ int add(DeviceVariableCollection&,int,int);		//add an element, -1 if fail
	__device__ int read(DeviceVariableCollection&,int);			//returns last and delete last element, -1 if fail
	__device__ int expansion(DeviceVariableCollection&, int);	//expansion as WorkSet

	__device__ void print();					//print
	__device__ void printLocks();
	__device__ int stillInQueue();

	__device__ ~DeviceParallelQueue();				//do nothing
};

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ DeviceParallelQueue::DeviceParallelQueue(){}

//////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ParallelQueueExternInit(DeviceVariableCollection* deviceVariableCollection,
									    DeviceVariable* deviceVariable, int* variablesMem,
									    int* lastValuesMem, int* lockReading, Triple* tripleQueueMem,
									    int nQueen, int nVariableCollection){

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < nQueen*nVariableCollection){

		deviceVariable[index].init2(&variablesMem[index*nQueen],nQueen);

		if(index < nVariableCollection){

			deviceVariableCollection[index].init2(&deviceVariable[index*nQueen],
												 &tripleQueueMem[index*nQueen*nQueen*3],
												 &variablesMem[index*nQueen*nQueen],
												 &lastValuesMem[index*nQueen],nQueen);

		}

	}

	if(index < nVariableCollection)
		lockReading[index] = 0;
}

__device__ DeviceParallelQueue::DeviceParallelQueue(DeviceVariableCollection* dvc, 
													DeviceVariable* dv,
													int* vm, int* lvm, int* lr, int* ll,
													Triple* tqm,
													int nq, int sz):
													deviceVariableCollection(dvc),deviceVariable(dv),
													variablesMem(vm),levelLeaved(ll),lastValuesMem(lvm),tripleQueueMem(tqm),
													lockReading(lr),nQueen(nq),size(sz),maxUsed(0),lockMaxUsed(0){

	ParallelQueueExternInit<<<int(size*nQueen)/1000+1,1000>>>(deviceVariableCollection,
															  deviceVariable,
															  variablesMem,
											    			  lastValuesMem,
											    			  lockReading,
															  tripleQueueMem,
															  nQueen,size);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceParallelQueue::DeviceParallelQueue::EXTERN INIT");


}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::init(DeviceVariableCollection* dvc, DeviceVariable* dv,
								 	  int* vm, int* lvm, int* lr, int* ll, Triple* tqm, int nq, int sz){

	variablesMem = vm;
	lastValuesMem = lvm;
	tripleQueueMem = tqm;

	deviceVariable = dv;
	deviceVariableCollection = dvc;

	lockReading = lr;
	levelLeaved = ll;

	nQueen = nq;
	size = sz;
	maxUsed = 0;
	lockMaxUsed = 0;

	ParallelQueueExternInit<<<int(size*nQueen)/1000+1,1000>>>(deviceVariableCollection,
											 				  deviceVariable,
											 				  variablesMem,
							    			 				  lastValuesMem,
							    			 				  lockReading,
											 				  tripleQueueMem,
											 				  nQueen,size);
	ErrorChecking::deviceErrorCheck(hipPeekAtLastError(),"DeviceParallelQueue::init::EXTERN INIT");
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::add(DeviceVariableCollection& element, int level, int index){

	int pos = -1;
	for (int i = 0; i < size; ++i){
		if(atomicCAS(&lockReading[i],0,1)==0){
			pos = i;
			break;
		}
	}

	if(pos == -1)return -1;

	while(atomicCAS(&lockMaxUsed,0,1)==1){}
	if(pos >= maxUsed)maxUsed = pos+1;
	lockMaxUsed = 0;

	levelLeaved[pos] = level;
	deviceVariableCollection[pos] = element;

	ErrorChecking::deviceErrorCheck(hipDeviceSynchronize(),"SYNCH");

	lockReading[pos] = 2;

	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::read(DeviceVariableCollection& element, int index){

	__shared__ int pos;
	pos = -1;

	__syncthreads();

	if(threadIdx.x == 0){
		for (int i = 0; i < maxUsed && i < size; ++i){
			if(atomicCAS(&lockReading[i],2,3)==2){
				pos = i;
				break;
			}
		}
	}

	__syncthreads();

	if(pos == -1)return -1;
	
	element = deviceVariableCollection[pos];

	int ltemp = levelLeaved[pos];

	__syncthreads();

	if(threadIdx.x == 0)lockReading[pos] = 0;

	return ltemp;
}


//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::expansion(DeviceVariableCollection& element, int level){


/*	if(level > nQueen || level < 0){
		if(threadIdx.x == 0)ErrorChecking::deviceError("Error::DeviceParallelQueue::expansion::LEVEL OUT OF BOUND");
		return -1;
	}*/


	DeviceQueenPropagation deviceQueenPropagation;
	if(nQueen > 20) return -1;
	__shared__ int first;
	__shared__ int nValues;
	__shared__ int positions[20];
	__shared__ int values[20];
	__shared__ bool ok;

	nValues = 0;
	first = -1;
	ok = false;

__syncthreads();

	for(int val = 0; val < nQueen; ++val){

		if(element.deviceVariable[level].domain[val] == 1 && first != -1){

			ok = false;

			__syncthreads();

			if(threadIdx.x == 0){

				for (int i = 0; i < size; ++i){

					if(atomicCAS(&lockReading[i],0,1)==0){

						positions[nValues] = i;
						values[nValues] = val;
						//printf("adding %d\n", positions[nValues]);
						ok = true;
						++nValues;
						break;

					}

				}
				
			}

			__syncthreads();

			if(!ok){
				//non sono riuscito a occupare per tutto il livello, tolgo i lock
				if(threadIdx.x == 0){
					for(int i = 0; i < nValues; ++i){
						lockReading[positions[i]] = 0;
					}
				}
				return -1;
			}

		}else if(element.deviceVariable[level].domain[val] == 1 && first == -1){
			__syncthreads();
			first = val;
		}
	}

	__syncthreads();


	for(int i = 0; i < nValues; ++i){

		deviceVariableCollection[positions[i]].deviceVariable[level].ground = values[i];

		if(threadIdx.x == 0)atomicMax(&maxUsed, positions[i]+1);

		levelLeaved[positions[i]] = level+1;

		deviceVariableCollection[positions[i]] = element;
		
		deviceVariableCollection[positions[i]].lastValues[level] = nQueen;


		if(threadIdx.x < nQueen && threadIdx.x != values[i]){
			--deviceVariableCollection[positions[i]].deviceVariable[level].domain[threadIdx.x];
		}

		deviceVariableCollection[positions[i]].deviceVariable[level].ground = values[i];

		__syncthreads();

		deviceQueenPropagation.parallelForwardChecking(deviceVariableCollection[positions[i]],level,values[i]);

		__syncthreads();

		if(threadIdx.x == 0)lockReading[positions[i]] = 2;

	}

	__syncthreads();

	int val = first;

	element.lastValues[level] = nQueen;
	element.deviceVariable[level].ground = val;

	if(threadIdx.x < nQueen && threadIdx.x != val){
		--element.deviceVariable[level].domain[threadIdx.x];
	}

	__syncthreads();

	deviceQueenPropagation.parallelForwardChecking(element,level,val);

	return nValues+1;
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::print(){

	int count = 0;
	for(int i = 0; i < size; ++i) {
		if(lockReading[i] != 0)printf("------[%d,%d,%d]------\n", i,lockReading[i],levelLeaved[i]);
		if(lockReading[i] != 0)deviceVariableCollection[i].print();
		if(lockReading[i] != 0)++count;
	}

	printf("count:%d \n",count);
	printf("size: %d\n",size);
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ void DeviceParallelQueue::printLocks(){
	for(int i = 0; i < size; ++i){
		if(i % 100 == 0)printf("\n");
		printf("%d", lockReading[i]);
	}printf("\n");
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ int DeviceParallelQueue::stillInQueue(){
	int sum = 0;
	for(int i = 0; i < size; ++i){
		if(lockReading[i] > 0)++sum;
	}
	return sum;
}

//////////////////////////////////////////////////////////////////////////////////////////////

__device__ DeviceParallelQueue::~DeviceParallelQueue(){}

//////////////////////////////////////////////////////////////////////////////////////////////
