#include "hip/hip_runtime.h"
#pragma once
#include "../Variable/Variable.cu"
#include "../TripleQueue/TripleQueue.cu"
#include "../ErrorChecking/ErrorChecking.cu"
#include "../MemoryManagement/MemoryManagement.cu"

///////////////////////////////////////////////////////////////////////
////////////////////////HOST SIDE//////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct HostVariableCollection{
	int* dMem;							//ptr to deviceMemory
	DeviceVariable* deviceVariableMem;	//vector for variables struct
	int* dMemlastValues;				//last values array
	int nQueen;							//number of variables and also domain size
	HostQueue hostQueue;				//queue

	__host__ HostVariableCollection(int);		//allocate memory with hostMemoryManagemnt
	__host__ ~HostVariableCollection();			//deallocate dMemVariables
};

///////////////////////////////////////////////////////////////////////

__host__ HostVariableCollection::HostVariableCollection(int nq):
	nQueen(nq),hostQueue(nq){

	ErrorChecking::hostMessage("Warn::HostVariableCollection::constructor::ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&deviceVariableMem,sizeof(DeviceVariable)*nQueen),"HostVariableCollection::HostVariableCollection::DEVICE VARIABLE ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&dMemlastValues,sizeof(int)*nQueen),"HostVariableCollection::HostVariableCollection::LAST VALUE ALLOCATION");
	ErrorChecking::hostErrorCheck(hipMalloc((void**)&dMem,sizeof(int)*nQueen*nQueen),"HostVariableCollection::HostVariableCollection::VARIABLE MEM ALLOCATION");
}

///////////////////////////////////////////////////////////////////////

__host__ HostVariableCollection::~HostVariableCollection(){
	ErrorChecking::hostMessage("Warn::HostVariableCollection::destructor::DELLOCATION");
	ErrorChecking::hostErrorCheck(hipFree(deviceVariableMem),"HostVariableCollection::~HostVariableCollection::DEVICE VARIABLE DEALLOCATION");;
	ErrorChecking::hostErrorCheck(hipFree(dMemlastValues),"HostVariableCollection::~HostVariableCollection::DEVICE VARIABLE DEALLOCATION");;
	ErrorChecking::hostErrorCheck(hipFree(dMem),"HostVariableCollection::~HostVariableCollection::DEVICE VARIABLE DEALLOCATION");;
}

///////////////////////////////////////////////////////////////////////
////////////////////////DEVICE SIDE////////////////////////////////////
///////////////////////////////////////////////////////////////////////

struct DeviceVariableCollection{

	int fullParallel;				//chose parallel code
	int nQueen;						//number of variables and domain size
	int* lastValues;				//last values array
	int* dMem;	
	DeviceVariable* deviceVariable;	//array for variables
	DeviceQueue deviceQueue;		//triple queue

	__device__ DeviceVariableCollection();											//do nothing
	__device__ DeviceVariableCollection(DeviceVariable*,Triple*, int*,int*,int);	//initialize
	__device__ void init(DeviceVariable*,Triple*,int*,int*,int);					//initialize
	__device__ void init2(DeviceVariable*,Triple*,int*,int*,int);					//initialize
	__device__ void init3(DeviceVariable*,Triple*,int*,int*,int);					//initialize
	__device__ ~DeviceVariableCollection();											//do nothing

	__device__ DeviceVariableCollection& operator=(DeviceVariableCollection&);			//copy

	__device__ bool isGround();			//check if every variable is not failed
	__device__ bool isFailed();			//check if every variable is ground

	__device__ void print();			//print collection

};

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::DeviceVariableCollection(){}

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::DeviceVariableCollection(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq):
	fullParallel(true),nQueen(nq),deviceVariable(dv),deviceQueue(q,nq),lastValues(lv),dMem(vm){
	
	for(int i = 0; i < nQueen*nQueen; ++i){
		vm[i] = 1;
	}

	for (int i = 0; i < nQueen; ++i){
		deviceVariable[i].init2(&vm[nQueen*i],nQueen);
		lastValues[i]=0;
	}

}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::init(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq){
	
	dMem = vm;
	fullParallel = true;
	nQueen = nq;
	deviceVariable = dv;
	lastValues = lv;
	deviceQueue.init(q,nq);

	if(threadIdx.x < nQueen*nQueen){
		vm[threadIdx.x] = 1;
	}

	if(threadIdx.x < nQueen){
		deviceVariable[threadIdx.x].init2(&vm[nQueen*threadIdx.x],nQueen);
		lastValues[threadIdx.x]=0;
	}

}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::init2(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq){

	fullParallel = true;
	dMem = vm;
	nQueen = nq;
	deviceVariable = dv;
	lastValues = lv;
	deviceQueue.init(q,nq);

	for (int i = 0; i < nQueen; ++i){
		deviceVariable[i].init2(&vm[nQueen*i],nQueen);
		lastValues[i]=0;
	}

}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::init3(DeviceVariable* dv,Triple* q, int* vm, int* lv, int nq){

	fullParallel = true;
	dMem = vm;
	nQueen = nq;
	deviceVariable = dv;
	lastValues = lv;
	deviceQueue.init(q,nq);

}

///////////////////////////////////////////////////////////////////////

__device__ DeviceVariableCollection::~DeviceVariableCollection(){}

///////////////////////////////////////////////////////////////////////

__device__ bool DeviceVariableCollection::isGround(){
	for(int i = 0; i < nQueen; ++i)
		if(deviceVariable[i].ground==-1)return false;

	return true;
}

///////////////////////////////////////////////////////////////////////

__device__ bool DeviceVariableCollection::isFailed(){
	for(int i = 0; i < nQueen; ++i)
		if(deviceVariable[i].failed == 1)return true;

	return false;
}

///////////////////////////////////////////////////////////////////////

__device__ void externCopy(DeviceVariableCollection& to,DeviceVariableCollection& other){

	__shared__ int nQueen; 
	__shared__ int next1; 
	__shared__ int next2; 
	__shared__ int next3;

	nQueen = to.nQueen;
	
	next1 = ((((int(3*nQueen*nQueen/32)+1)*32)-3*nQueen*nQueen)+3*nQueen*nQueen);
	next2 = ((((int((next1+nQueen*nQueen)/32)+1)*32)-(next1+nQueen*nQueen))+(next1+nQueen*nQueen));
	next3 = ((((int((next2+nQueen)/32)+1)*32)-(next2+nQueen))+(next2+nQueen));

	if(threadIdx.x < 3*nQueen*nQueen)
		to.deviceQueue.q[threadIdx.x] = other.deviceQueue.q[threadIdx.x];

	if(threadIdx.x >=  next1 && threadIdx.x < next1 + nQueen*nQueen)
		to.dMem[threadIdx.x - next1] = other.dMem[threadIdx.x - next1];

	if(threadIdx.x >= next2 && threadIdx.x < next2 + nQueen)
		to.lastValues[threadIdx.x - next2] = other.lastValues[threadIdx.x- next2];

	if(threadIdx.x >= next3 && threadIdx.x < next3 + nQueen){
		to.deviceVariable[threadIdx.x - next3].ground = other.deviceVariable[threadIdx.x - next3].ground;
		to.deviceVariable[threadIdx.x - next3].failed = other.deviceVariable[threadIdx.x - next3].failed;
		to.deviceVariable[threadIdx.x - next3].changed = other.deviceVariable[threadIdx.x - next3].changed;
	}

	if(threadIdx.x == 1023)
		to.deviceQueue.count = other.deviceQueue.count;

}

__device__ DeviceVariableCollection& DeviceVariableCollection::operator=(DeviceVariableCollection& other){

/*	__shared__ int next1; 
	__shared__ int next2; 
	__shared__ int next3;

	next1 = ((((int(3*nQueen*nQueen/32)+1)*32)-3*nQueen*nQueen)+3*nQueen*nQueen);
	next2 = ((((int((next1+nQueen*nQueen)/32)+1)*32)-(next1+nQueen*nQueen))+(next1+nQueen*nQueen));
	next3 = ((((int((next2+nQueen)/32)+1)*32)-(next2+nQueen))+(next2+nQueen));

	if(threadIdx.x < 3*nQueen*nQueen)
		this->deviceQueue.q[threadIdx.x] = other.deviceQueue.q[threadIdx.x];

	if(threadIdx.x >=  next1 && threadIdx.x < next1 + nQueen*nQueen)
		this->dMem[threadIdx.x - next1] = other.dMem[threadIdx.x - next1];

	if(threadIdx.x >= next2 && threadIdx.x < next2 + nQueen)
		this->lastValues[threadIdx.x - next2] = other.lastValues[threadIdx.x- next2];

	if(threadIdx.x >= next3 && threadIdx.x < next3 + nQueen){
		this->deviceVariable[threadIdx.x - next3].ground = other.deviceVariable[threadIdx.x - next3].ground;
		this->deviceVariable[threadIdx.x - next3].failed = other.deviceVariable[threadIdx.x - next3].failed;
		this->deviceVariable[threadIdx.x - next3].changed = other.deviceVariable[threadIdx.x - next3].changed;
	}

	if(threadIdx.x == 1023)
		this->deviceQueue.count = other.deviceQueue.count;*/

	if(threadIdx.x < 3*nQueen*nQueen)
		this->deviceQueue.q[threadIdx.x] = other.deviceQueue.q[threadIdx.x];

	if(threadIdx.x < nQueen*nQueen)
		this->dMem[threadIdx.x] = other.dMem[threadIdx.x];

	if(threadIdx.x < nQueen)
		this->lastValues[threadIdx.x] = other.lastValues[threadIdx.x];

	if(threadIdx.x < nQueen){
		this->deviceVariable[threadIdx.x].ground = other.deviceVariable[threadIdx.x].ground;
		this->deviceVariable[threadIdx.x].failed = other.deviceVariable[threadIdx.x].failed;
		this->deviceVariable[threadIdx.x].changed = other.deviceVariable[threadIdx.x].changed;
	}

	if(threadIdx.x == 1023)
		this->deviceQueue.count = other.deviceQueue.count;

	return *this;
}

///////////////////////////////////////////////////////////////////////

__device__ void DeviceVariableCollection::print(){
	for (int i = 0; i < nQueen; ++i){
		printf("[%d] ::: ",lastValues[i]);
		deviceVariable[i].print();
	}
	deviceQueue.print();
	printf("\n");
}

///////////////////////////////////////////////////////////////////////
