#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void externUndoAssignParallel(int* domain, int size, int value){

if(threadIdx.x + blockIdx.x * blockDim.x < size &&
threadIdx.x + blockIdx.x * blockDim.x != value)
++domain[threadIdx.x + blockIdx.x * blockDim.x];

}