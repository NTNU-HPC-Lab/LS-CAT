#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/*	add_up
*	Función del kernel para sumar hacia arriba todos los números que sean iguales.
*/
__device__ void stack_up(int *matriz, int anchura, int altura, int x, int y) {
for (int i = altura - 1; i > 0; i--)  //realizaremos el desplazamiento celda a celda una altura-1 veces para gestionar la posibilidad del ultimo poniendose el primero de la lista
{
if ((x != 0) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y - anchura)] == 0) //Si la celda pertenece a la primera fila, es 0 o su superior no es 0, no hace nada
{
matriz[x*anchura + (y - anchura)] = matriz[x*anchura + y]; //Si lo es, desplazamos la celda
matriz[x*anchura + y] = 0;
}
__syncthreads(); //utilizamos una sincronizacion para que estos pasos sean realizados a la vez por los hilos del bloque y
}
}
__device__ void add_up(int *matriz, int x, int y, int altura, int anchura)
{
if (x != 0 && y < anchura) //Los primeros hilos no deben realizar ninguna operacion pues serán modificados por los demas
{
if (matriz[x*anchura + y] != 0) //Si es distinto de 0, gestiona su posible suma o desplazamiento
{
if (matriz[x*anchura + y] == matriz[(x-1)*anchura + y]) //Si es igual a su superior, se procede a comprobar el numero de celdas con el mismo numero que hay en esa columna
{
int iguales = 0;
iguales++;
for (int i = 1; i <= x; i++)
{
if (matriz[x*anchura + y] == matriz[(x-i)*anchura + y])
{
iguales++;
}
else {
break;
}
}
if (iguales % 2 == 0) //Si el numero es par, se suman, si no, ese numero será mezclado con otro y no estará disponible
{
matriz[(x-1)*anchura + y] = matriz[(x-1)*anchura + y] * 2;
matriz[x*anchura + y] = 0;
}
}
else if (matriz[(x - 1)*anchura + y] == 0) //Se comprueba que otros hilos hayan dejado 0 en sus operaciones para desplazarse
{
matriz[(x - 1)*anchura + y] = matriz[x*anchura + y];
matriz[x*anchura + y] = 0;
}
}
}
}
__global__ void mov_upK(int *matriz, int anchura, int altura) {
int x = threadIdx.x;
int y = threadIdx.y;

stack_up(matriz, anchura, altura, x, y); //Realizamos las llamadas de la siguiente manera para gestionar el movimiento:
add_up(matriz, x, y, altura, anchura); //2 2 0 4   -> 4 4 0 0
__syncthreads();
stack_up(matriz, anchura, altura, x, y);
}