#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/*	add_up
*	Función del kernel para sumar hacia arriba todos los números que sean iguales.
*/
__device__ void stack_left(int *matriz, int anchura, int altura, int x, int y) {

for (int i = anchura-1; i > 0; i--) //realizaremos el desplazamiento celda a celda una altura-1 veces para gestionar la posibilidad del ultimo poniendose el primero de la lista
{
if ( (y != 0) && (matriz[x*anchura +y]!=0) && matriz[x*anchura + (y - 1)] == 0) //Si la celda pertenece a la primera fila, es 0 o su superior no es 0, no hace nada
{
matriz[x*anchura + (y - 1)] = matriz[x*anchura + y]; //Si lo es, desplazamos la celda
matriz[x*anchura + y] = 0;
}
__syncthreads(); //utilizamos una sincronizacion para que estos pasos sean realizados a la vez por los hilos del bloque
}
}
__device__ void add_left(int *matriz, int x, int y, int altura, int anchura)
{
if (y != 0 && y < anchura) //Los primeros hilos de la izquierda no deben realizar ninguna operacion pues serán modificados por los demas
{
if (matriz[x*anchura + y] != 0) //Si es distinto de 0, gestiona su posible suma o desplazamiento
{
if (matriz[x*anchura + y] == matriz[x*anchura + (y - 1)]) //Si es igual a su vecino izquierdo, se procede a comprobar el numero de celdas con el mismo numero que hay en esa columna
{
int iguales = 0;
iguales++;
for (int i = 1; i <= y; i++)
{
if (matriz[x*anchura + y] == matriz[x*anchura + (y - i)])
{
iguales++;
}
else {
break;
}
}
if (iguales % 2 == 0) //Si el numero es par, se suman, si no, ese numero será mezclado con otro y no estará disponible
{
matriz[x*anchura + (y - 1)] = matriz[x*anchura + (y - 1)] * 2;
matriz[x*anchura + y] = 0;
}
}
else if (matriz[x*anchura + (y - 1)] == 0) //Se comprueba que otros hilos hayan dejado 0 en sus operaciones para desplazarse
{
matriz[x*anchura + (y - 1)] = matriz[x*anchura + y];
matriz[x*anchura + y] = 0;
}
}
}
}
__global__ void mov_leftK(int *matriz, int anchura, int altura) {
int x = threadIdx.x;
int y = threadIdx.y;

stack_left(matriz, anchura, altura, x, y); // Realizamos las llamadas de la siguiente manera para gestionar el movimiento :
add_left(matriz, x, y, altura, anchura);   //2 2 0 4   -> 4 4 0 0
__syncthreads();
stack_left(matriz, anchura, altura, x, y);
}