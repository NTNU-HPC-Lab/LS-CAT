#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/*	add_up
*	Función del kernel para sumar hacia arriba todos los números que sean iguales.
*/
__device__ void stack_right(int *matriz, int anchura, int altura, int x, int y)
{

for (int i = anchura - 1; i > 0; i--) //realizaremos el desplazamiento celda a celda una altura-1 veces para gestionar la posibilidad del ultimo poniendose el primero de la lista
{
if ((y != anchura-1) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y + 1)] == 0) //Si la celda pertenece a la primera fila, es 0 o su superior no es 0, no hace nada
{
matriz[x*anchura + (y + 1)] = matriz[x*anchura + y]; //Si lo es, desplazamos la celda
matriz[x*anchura + y] = 0;
}
__syncthreads(); //utilizamos una sincronizacion para que estos pasos sean realizados a la vez por los hilos del bloque
}
}
__device__ void add_right(int *matriz, int x, int y, int altura, int anchura)
{
if (y != anchura-1 && y < anchura) //Los primeros hilos de la derecha no deben realizar ninguna operacion pues serán modificados por los demas
{
if (matriz[x*anchura + y] != 0) //Si es distinto de 0, gestiona su posible suma o desplazamiento
{
if (matriz[x*anchura + y] == matriz[x*anchura + (y + 1)]) //Si es igual a su superior, se procede a comprobar el numero de celdas con el mismo numero que hay en esa columna

{
int iguales = 0;
iguales++;
for (int i = 1; y + i < anchura; i++)
{
if (matriz[x*anchura + y] == matriz[x*anchura + (y + i)])
{
iguales++;
}
else {
break;
}
}
if (iguales % 2 == 0) //Si el numero es par, se suman, si no, ese numero será mezclado con otro y no estará disponible
{
matriz[x*anchura + (y + 1)] = matriz[x*anchura + (y + 1)] * 2;
matriz[x*anchura + y] = 0;
}
}
else if (matriz[x*anchura + (y + 1)] == 0) // Se comprueba que otros hilos hayan dejado 0 en sus operaciones para desplazarse
{
matriz[x*anchura + (y + 1)] = matriz[x*anchura + y];
matriz[x*anchura + y] = 0;
}
}
}
}
__global__ void mov_rightK(int *matriz, int anchura, int altura)
{
int x = threadIdx.x;
int y = threadIdx.y;

stack_right(matriz, anchura, altura, x, y); //Realizamos las llamadas de la siguiente manera para gestionar el movimiento:
add_right(matriz, x, y, altura, anchura); //2 2 0 4   -> 4 4 0 0
__syncthreads();
stack_right(matriz, anchura, altura, x, y);
}