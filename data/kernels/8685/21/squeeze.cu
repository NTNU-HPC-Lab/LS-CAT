#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squeeze(float *B, int dim, int length, float L, float M) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
if (index < length + dim) {
B[index] = 1 / (1 + expf(-1 * L * (B[index] - M)));
}
}