#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateState(float *B, float *external, int dim, float timestep, int length, float L, float M) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
if (index < length + dim) {
float input = B[index] + external[index];
float old_output = B[index - dim];
float d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * L * (input - M)));
B[index] = old_output + d_layers * timestep;
}
}