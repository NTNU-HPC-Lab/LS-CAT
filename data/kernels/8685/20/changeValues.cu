#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(float *matrix, int size) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x;
if (index < size) {
float a = matrix[index] * 10;
int b = (int) a;
matrix[index] = (float) b;

}
}