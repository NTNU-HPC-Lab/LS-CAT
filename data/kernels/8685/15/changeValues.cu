#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(double *matrix, int size) {
int index = (blockIdx.x * blockDim.x) + threadIdx.x;
if (index < size) {
double a = matrix[index] * 10;
int b = (int) a;
matrix[index] = (double) b;

}
}