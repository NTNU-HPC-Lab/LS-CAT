#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void integrateBinsT(int width, int height, int nbins, int binPitch, int* devIntegrals) {
const int blockY = blockDim.y * blockIdx.x;
const int threadY = threadIdx.y;
const int bin = threadIdx.x;
const int y = blockY + threadY;
if (y >= height) return;
if (bin >= binPitch) return;
int* imagePointer = devIntegrals + binPitch * y * width + bin;
int accumulant = 0;
for(int x = 0; x < width; x++) {
accumulant += *imagePointer;
*imagePointer = accumulant;
imagePointer += binPitch;
}
}