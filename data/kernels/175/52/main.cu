#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "findCentroidsAtomicFreeLocal_64.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int afLocal = 1;
int *responses = NULL;
hipMalloc(&responses, XSIZE*YSIZE);
int nPixels = 1;
int *cluster = NULL;
hipMalloc(&cluster, XSIZE*YSIZE);
int *centroidMass = NULL;
hipMalloc(&centroidMass, XSIZE*YSIZE);
unsigned int *centroidCount = NULL;
hipMalloc(&centroidCount, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
findCentroidsAtomicFreeLocal_64<<<gridBlock,threadBlock>>>(afLocal,responses,nPixels,cluster,centroidMass,centroidCount);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
findCentroidsAtomicFreeLocal_64<<<gridBlock,threadBlock>>>(afLocal,responses,nPixels,cluster,centroidMass,centroidCount);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
findCentroidsAtomicFreeLocal_64<<<gridBlock,threadBlock>>>(afLocal,responses,nPixels,cluster,centroidMass,centroidCount);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}