#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=


#define XBLOCK 16
#define YBLOCK 16





__global__ void normalizeLab_kernel(uint width, uint height, float* devL, float* devA, float* devB) {
int x0 = blockDim.x * blockIdx.x + threadIdx.x;
int y0 = blockDim.y * blockIdx.y + threadIdx.y;
if ((x0 < width) && (y0 < height)) {
int index = y0 * width + x0;
const float ab_min = -73;
const float ab_max = 95;
const float ab_range = ab_max - ab_min;
/* normalize Lab image */
float l_val = devL[index] / 100.0f;
float a_val = (devA[index] - ab_min) / ab_range;
float b_val = (devB[index] - ab_min) / ab_range;
if (l_val < 0) { l_val = 0; } else if (l_val > 1) { l_val = 1; }
if (a_val < 0) { a_val = 0; } else if (a_val > 1) { a_val = 1; }
if (b_val < 0) { b_val = 0; } else if (b_val > 1) { b_val = 1; }
devL[index] = l_val;
devA[index] = a_val;
devB[index] = b_val;
}
}