#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;
if ((centroidNumber < 64) && (dimensionNumber < 34)) {
float totalCount = (float)centroidCount[centroidNumber];
float mass = (float)centroidMass[dimensionNumber * 64 + centroidNumber];
centroids[dimensionNumber * 64 + centroidNumber] = mass / ((float)INTCONFACTOR * totalCount);
}
}