#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void combine_kernel(int nPixels, int cuePitchInFloats, float* devBg, float* devCga, float* devCgb, float* devTg, float* devMpb, float* devCombinedg) {
int index = blockDim.x * blockIdx.x + threadIdx.x;
int orientation = threadIdx.y;
int orientedIndex = orientation * cuePitchInFloats + index;
if (index < nPixels) {
float accumulant = 0.0;
float accumulant2=0.0;
float* pointer = &devBg[orientedIndex];
accumulant += *pointer * coefficients[0];
accumulant2 += *pointer * weights[0];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[1];
accumulant2 += *pointer * weights[1];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[2];
accumulant2 += *pointer * weights[2];
pointer = &devCga[orientedIndex];
accumulant += *pointer * coefficients[3];
accumulant2 += *pointer * weights[3];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[4];
accumulant2 += *pointer * weights[4];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[5];
accumulant2 += *pointer * weights[5];
pointer = &devCgb[orientedIndex];
accumulant += *pointer * coefficients[6];
accumulant2 += *pointer * weights[6];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[7];
accumulant2 += *pointer * weights[7];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[8];
accumulant2 += *pointer * weights[8];
pointer = &devTg[orientedIndex];
accumulant += *pointer * coefficients[9];
accumulant2 += *pointer * weights[9];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[10];
accumulant2 += *pointer * weights[10];
pointer += 8 * cuePitchInFloats;
accumulant += *pointer * coefficients[11];
accumulant2 += *pointer * weights[11];
devMpb[orientedIndex] = accumulant;
devCombinedg[orientedIndex] = accumulant2;
}
}