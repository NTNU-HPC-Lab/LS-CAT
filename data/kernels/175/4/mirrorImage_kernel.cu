#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=


#define XBLOCK 16
#define YBLOCK 16





__global__ void mirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, int* devInput, int* devOutput) {
int x0 = blockDim.x * blockIdx.x + threadIdx.x;
int y0 = blockDim.y * blockIdx.y + threadIdx.y;
if ((x0 < borderWidth) && (y0 < borderHeight)) {
int x1 = 0;
int y1 = 0;
if (x0 < border) {
x1 = border - x0 - 1;
} else if (x0 < border + width) {
x1 = x0 - border;
} else {
x1 = border + 2 * width - x0 - 1;
}
if (y0 < border) {
y1 = border - y0 - 1;
} else if (y0 < border + height) {
y1 = y0 - border;
} else {
y1 = border + 2 * height - y0 - 1;
}
devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
}
}