#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillImage(int width, int height, int value, int* devOutput) {
int x = blockDim.x * blockIdx.x + threadIdx.x;
int y = blockDim.y * blockIdx.y + threadIdx.y;
int index = y * width + x;
if ((y < height) && (x < width)) {
devOutput[index] = value;
}
}