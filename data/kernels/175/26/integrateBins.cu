#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void integrateBins(int width, int height, int nbins, int* devImage, int binPitch, int* devIntegrals) {
__shared__ int pixels[16];
const int blockX = blockDim.y * blockIdx.x;
const int threadX = threadIdx.y;
const int bin = threadIdx.x;
const int x = blockX + threadX;
if (x >= width) return;
if (bin > nbins) return;
int* imagePointer = devImage + x;
int* outputPointer = devIntegrals + binPitch * x + bin;
int accumulant = 0;
for(int y = 0; y < height; y++) {
if (bin == 0) {
pixels[threadX] = *imagePointer;
}
__syncthreads();
if (pixels[threadX] == bin) accumulant++;
*outputPointer = accumulant;
imagePointer += width;
outputPointer += width * binPitch;
}
}