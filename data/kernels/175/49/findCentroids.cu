#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
__shared__ int localMasses[32*17];
__shared__ unsigned int localCounts[32];
int pixel = blockDim.x * blockIdx.x + threadIdx.x;
if (threadIdx.x < 32) {
for (int i = 0; i < 17; i++) {
localMasses[32 * i + threadIdx.x] = 0;
}
localCounts[threadIdx.x] = 0;
}
__syncthreads();
if (pixel < nPixels) {
int myCluster = cluster[pixel];
int myIndex = pixel;
for(int filter = 0; filter < 17; filter++) {
int myElement = responses[myIndex];
atomicAdd(localMasses + filter * 32 + myCluster, myElement);
myIndex += nPixels;
}
}
__syncthreads();
if (threadIdx.x < 32) {
for (int filter = 0; filter < 17; filter++) {
atomicAdd(centroidMass + filter * 32 + threadIdx.x, localMasses[threadIdx.x + filter * 32]);
localMasses[threadIdx.x + filter * 32] = 0;
}
}
__syncthreads();
if (pixel < nPixels) {
int myCluster = cluster[pixel];
// yunsup fixed
int myIndex = pixel + nPixels*17;
for(int filter = 0; filter < 17; filter++) {
int myElement = responses[myIndex];
atomicAdd(localMasses + filter * 32 + myCluster, myElement);
myIndex += nPixels;
}
atomicInc(localCounts + myCluster, 100000000);
}
__syncthreads();
if (threadIdx.x < 32) {
for (int filter = 17; filter < 34; filter++) {
atomicAdd(centroidMass + filter * 32 + threadIdx.x, localMasses[threadIdx.x + (filter - 17) * 32]);
}
atomicAdd(centroidCount + threadIdx.x, localCounts[threadIdx.x]);
}
}