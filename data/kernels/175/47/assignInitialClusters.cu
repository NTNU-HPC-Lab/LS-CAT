#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
int x = blockDim.x * blockIdx.x + threadIdx.x;
int y = blockDim.y * blockIdx.y + threadIdx.y;
int pixel = y * width + x;
if ((x < width) && (y < height)) {
int xBlock = x / ((width - 1) / 6 + 1);
int yBlock = y / ((height - 1) / 6 + 1);
int assignedCluster = yBlock * 6 + xBlock;

if (assignedCluster >= 32)
{
assignedCluster = 31;
}

cluster[y * width + x] = assignedCluster;
for(int i = 0; i < filterCount; i++) {
int index = pixel + i * nPixels;
int response = (int)(INTCONFACTOR * responses[index]);
intResponses[index] = response;
}
}
}