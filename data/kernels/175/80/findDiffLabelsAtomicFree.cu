#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {


int x = blockDim.x * blockIdx.x + threadIdx.x;
if (x < nPoints) {
int index = x;
float minDistance = 10000000;
int minCluster = -1;
for(int cluster = 0; cluster < nClusters; cluster++) {
float clusterDistance = devDiff[index];
if (clusterDistance < minDistance) {
minDistance = clusterDistance;
minCluster = cluster;
}
index += diffPitchInFloats;
}
int previousCluster = devClusters[x];
devClusters[x] = minCluster;
if (minCluster != previousCluster) {
//int change=*devChanges;
//change++;
//*devChanges = change;
}
}
}