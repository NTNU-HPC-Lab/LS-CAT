#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findLabels(int nPixels, int filterCount, int clusterCount, float* responses, float* centroids, int* clusters, int* changes) {
__shared__ float sharedCentroids[34 * 32];
__shared__ unsigned int localChanges;
int x = blockDim.x * blockIdx.x + threadIdx.x;
if (threadIdx.x < 32) {
for(int i = 0; i < 34; i++) {
float element = centroids[i * 64 + threadIdx.x];
sharedCentroids[i * 32 + threadIdx.x] = element;
}
}
__syncthreads();
int bestLabel = -1;
float bestDistance = 1000000;
if (x < nPixels) {
for(int label = 0; label < 32; label++) {
float accumulant = 0.0f;
int index = x;
for(int dimension = 0; dimension < 34; dimension++) {
float diff = sharedCentroids[dimension * 32 + label] - responses[index];
accumulant += diff * diff;
index += nPixels;
}
if (accumulant < bestDistance) {
bestLabel = label;
bestDistance = accumulant;
}
}
}
__syncthreads();
if (threadIdx.x < 32) {
for(int i = 0; i < 34; i++) {
sharedCentroids[i * 32 + threadIdx.x] = centroids[i * 64 + threadIdx.x + 32];
}
}
__syncthreads();

if (x < nPixels) {
for(int label = 0; label < 32; label++) {
float accumulant = 0.0f;
int index = x;
for(int dimension = 0; dimension < 34; dimension++) {
float diff = sharedCentroids[dimension * 32 + label] - responses[index];
accumulant += diff * diff;
index += nPixels;
}
if (accumulant < bestDistance) {
bestLabel = label + 32;
bestDistance = accumulant;
}
}
int formerCluster = clusters[x];
if (bestLabel != formerCluster) {
atomicInc(&localChanges, 10000000);
}
clusters[x] = bestLabel;
}
__syncthreads();
if (threadIdx.x == 0) {
atomicAdd(changes, localChanges);
}
}