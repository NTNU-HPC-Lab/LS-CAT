#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;
if ((centroidNumber < 32) && (dimensionNumber < 34)) {
float totalCount = (float)centroidCount[centroidNumber];
float mass = (float)centroidMass[dimensionNumber * 32 + centroidNumber];
centroids[dimensionNumber * 32 + centroidNumber] = mass / ((float)INTCONFACTOR * totalCount);
}
}