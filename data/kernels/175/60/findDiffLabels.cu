#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findDiffLabels(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
int x = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ unsigned int localChanges;
if (x < nPoints) {
int index = x;
float minDistance = 10000000;
int minCluster = -1;
for(int cluster = 0; cluster < nClusters; cluster++) {
float clusterDistance = devDiff[index];
if (clusterDistance < minDistance) {
minDistance = clusterDistance;
minCluster = cluster;
}
index += diffPitchInFloats;
}
int previousCluster = devClusters[x];
devClusters[x] = minCluster;
if (minCluster != previousCluster) {
atomicInc(&localChanges, 10000000);
}
}
__syncthreads();
if (threadIdx.x == 0) {
atomicAdd(devChanges, localChanges);
}
}