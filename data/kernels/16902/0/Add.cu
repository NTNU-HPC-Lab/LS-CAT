#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// GPU Code
// __global__ indicates that it is a GPU kernel, that can be called from the CPU

// CPU Code
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id < N)

d_c[id] = d_a[id] + d_b[id];

}