#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if(id < N)

d_c[id] = d_a[id] + d_b[id];

}