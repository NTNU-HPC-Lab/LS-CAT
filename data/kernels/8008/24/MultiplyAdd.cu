#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
int p = __mul24(y, width) + x;
if (x<width && y<height)
d_Result[p] = d_ConstantA[0]*d_Data[p] + d_ConstantB[0];
__syncthreads();
}