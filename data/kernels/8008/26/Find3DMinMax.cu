#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
//Data cache
__shared__ float data1[3*(MINMAX_SIZE + 2)];
__shared__ float data2[3*(MINMAX_SIZE + 2)];
__shared__ float data3[3*(MINMAX_SIZE + 2)];
__shared__ float ymin1[(MINMAX_SIZE + 2)];
__shared__ float ymin2[(MINMAX_SIZE + 2)];
__shared__ float ymin3[(MINMAX_SIZE + 2)];
__shared__ float ymax1[(MINMAX_SIZE + 2)];
__shared__ float ymax2[(MINMAX_SIZE + 2)];
__shared__ float ymax3[(MINMAX_SIZE + 2)];

//Current tile and apron limits, relative to row start
const int tx = threadIdx.x;
const int xStart = __mul24(blockIdx.x, MINMAX_SIZE);
const int xEnd = xStart + MINMAX_SIZE - 1;
const int xReadPos = xStart + tx - WARP_SIZE;
const int xWritePos = xStart + tx;
const int xEndClamped = min(xEnd, width - 1);
int memWid = MINMAX_SIZE + 2;

int memPos0 = (tx - WARP_SIZE + 1);
int memPos1 = (tx - WARP_SIZE + 1);
int yq = 0;
unsigned int output = 0;
for (int y=0;y<34;y++) {

output >>= 1;
int memPos =  yq*memWid + (tx - WARP_SIZE + 1);
int yp = 32*blockIdx.y + y - 1;
yp = max(yp, 0);
yp = min(yp, height-1);
int readStart = __mul24(yp, width);

//Set the entire data cache contents
if (tx>=(WARP_SIZE-1)) {
if (xReadPos<0) {
data1[memPos] = 0;
data2[memPos] = 0;
data3[memPos] = 0;
} else if (xReadPos>=width) {
data1[memPos] = 0;
data2[memPos] = 0;
data3[memPos] = 0;
} else {
data1[memPos] = d_Data1[readStart + xReadPos];
data2[memPos] = d_Data2[readStart + xReadPos];
data3[memPos] = d_Data3[readStart + xReadPos];
//if ((readStart + xReadPos)<0 || (readStart + xReadPos)>=width*height)
//  printf("Find3DMinMax: read error\n");
}
}
__syncthreads();

int memPos2 = yq*memWid + tx;
if (y>1) {
if (tx<memWid) {
float min1 = fminf(fminf(data1[memPos0], data1[memPos1]),
data1[memPos2]);
float min2 = fminf(fminf(data2[memPos0], data2[memPos1]),
data2[memPos2]);
float min3 = fminf(fminf(data3[memPos0], data3[memPos1]),
data3[memPos2]);
float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]),
data1[memPos2]);
float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]),
data2[memPos2]);
float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]),
data3[memPos2]);
ymin1[tx] = min1;
ymin2[tx] = fminf(fminf(min1, min2), min3);
ymin3[tx] = min3;
ymax1[tx] = max1;
ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
ymax3[tx] = max3;
}
}
__syncthreads();

if (y>1) {
if (tx<MINMAX_SIZE) {
if (xWritePos<=xEndClamped) {
float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]),
ymin1[tx+1]), ymin3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
minv = fminf(minv, d_ConstantA[1]);
float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]),
ymax1[tx+1]), ymax3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
maxv = fmaxf(maxv, d_ConstantA[0]);
if (data2[memPos1+1]<minv || data2[memPos1+1]>maxv)
output |= 0x80000000;
}
}
}
__syncthreads();

memPos0 = memPos1;
memPos1 = memPos2;
yq = (yq<2 ? yq+1 : 0);
}
if (tx<MINMAX_SIZE && xWritePos<width) {
int writeStart = __mul24(blockIdx.y, width) + xWritePos;
d_Result[writeStart] = output;
//if (writeStart<0 || writeStart>=width*iDivUp(height,32))
//  printf("Find3DMinMax: write error\n");
}
}