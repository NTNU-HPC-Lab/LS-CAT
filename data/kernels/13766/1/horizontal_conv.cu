#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

struct  pixel //to store RGB values
{
unsigned char r;
unsigned char g;
unsigned char b;
};

__device__ pixel padding(pixel* Pixel_val, int x_coord, int y_coord, int img_width, int img_height)
{	pixel Px;
Px.r=0; Px.g=0; Px.b=0;
if(x_coord< img_width && y_coord <img_height && x_coord>=0 && y_coord>=0)
{
Px=Pixel_val[y_coord*img_width+x_coord];
}
return Px;
}
__global__ void horizontal_conv(pixel* Pixel_in, pixel* Pixel_out, int img_wd, int img_ht, float* kernel, int k)
{
float tmp_r, tmp_b, tmp_g;
//horizontal convolution
//int pix_idx=blockIdx.x*blockDim.x + threadIdx.x;
//int row=(int)(pix_idx/img_wd);
//int col=pix_idx%img_wd;
size_t col=blockIdx.x*blockDim.x + threadIdx.x;
size_t row=blockIdx.y*blockDim.y + threadIdx.y;
size_t pix_idx=row*img_wd+col;

tmp_r=0, tmp_g=0, tmp_b=0;
if(row<img_ht && col<img_wd)
{
for(int l=0; l<k;l++)
{
pixel pix_val=padding(Pixel_in, col+ l-(k-1)/2, row, img_wd, img_ht);
tmp_r+=pix_val.r * kernel[l];
tmp_g+=pix_val.g * kernel[l];
tmp_b+=pix_val.b * kernel[l];
}
Pixel_out[pix_idx].r=tmp_r;
Pixel_out[pix_idx].g=tmp_g;
Pixel_out[pix_idx].b=tmp_b;
}
}