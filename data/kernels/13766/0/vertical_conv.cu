#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

struct  pixel //to store RGB values
{
unsigned char r;
unsigned char g;
unsigned char b;
};

__device__ pixel padding(pixel* Pixel_val, int x_coord, int y_coord, int img_width, int img_height)
{	pixel Px;
Px.r=0; Px.g=0; Px.b=0;
if(x_coord< img_width && y_coord <img_height && x_coord>=0 && y_coord>=0)
{
Px=Pixel_val[y_coord*img_width+x_coord];
}
return Px;
}
__global__ void vertical_conv(pixel* Pixel_in_v, pixel* Pixel_out_v,int img_wd_v, int img_ht_v, float* kernel_v, int k_v)
{
float tmp_r, tmp_g, tmp_b;
//int pix_idx_v=blockIdx.x*blockDim.x + threadIdx.x;
//int row=(int)(pix_idx_v/img_wd_v);
//int col=pix_idx_v%img_wd_v;
size_t col=blockIdx.x*blockDim.x + threadIdx.x;
size_t row=blockIdx.y*blockDim.y + threadIdx.y;
size_t pix_idx_v=row*img_wd_v+col;
tmp_r=0, tmp_g=0, tmp_b=0;
if(row<img_ht_v && col<img_wd_v){

for(int l=0;l<k_v;l++)
{//doing by 1 D arrays
pixel pix_val=padding(Pixel_in_v, col, (row+l-(k_v-1)/2), img_wd_v, img_ht_v);
tmp_r+=pix_val.r * kernel_v[l];
tmp_b+=pix_val.b * kernel_v[l];
tmp_g+=pix_val.g * kernel_v[l];
}

Pixel_out_v[pix_idx_v].r=tmp_r;
Pixel_out_v[pix_idx_v].g=tmp_g;
Pixel_out_v[pix_idx_v].b=tmp_b;
}
}