#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cubic(float* d_out, float* d_in)
{
int idx = threadIdx.x;
float f = d_in[idx];
d_out[idx] = f * f * f;
}