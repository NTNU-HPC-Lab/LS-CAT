#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(float* d_out, float* d_in)
{
int idx = threadIdx.x;   // here depends on the <<<block, threadPerBlock>>>,  build-in variable: threadIdx
float f = d_in[idx];
d_out[idx] = f * f;
}