#include "hip/hip_runtime.h"
#include "includes.h"
//Example 5.2.1. Pg 67, multiple block/threads
#define N (33 * 1024)



__global__ void add(int *a, int *b, int *c) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < N) {
c[tid] = a[tid] + b[tid];
tid += blockDim.x * gridDim.x;
}
}