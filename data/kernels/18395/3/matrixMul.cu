#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(float *M, float *N, float *P, int width)
{
int col= blockDim.x * blockIdx.x + threadIdx.x;
int row = blockDim.y * blockIdx.y + threadIdx.y;
if (row < width && col < width)
{
float pValue = 0;
for(int k=0; k<width; k++)
pValue += M[row * width + k] * N[k * width + col];
P[row * width + col] = pValue;
}
}