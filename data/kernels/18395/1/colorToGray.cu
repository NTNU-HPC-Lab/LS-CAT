#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorToGray(unsigned char *input, unsigned char *output, int height, int width)
{
int col= blockDim.x * blockIdx.x + threadIdx.x;
int row = blockDim.y * blockIdx.y + threadIdx.y;
float scale[3] = {0.299, 0.587, 0.114};
if (row < height && col < width)
{
int pixelIndex = row * width + col;
int rgbIndex = pixelIndex * 3;

unsigned char r = input[rgbIndex];                 // rgb rgb rgb rgb rgb
unsigned char g = input[rgbIndex + 1];
unsigned char b = input[rgbIndex + 2];
output[pixelIndex] = r* scale[0] + g * scale[1] + b*scale[2];
}
}