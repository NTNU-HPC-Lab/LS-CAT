#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void meanFilter(unsigned char *input, unsigned char *output, int height, int width)
{
int col= blockDim.x * blockIdx.x + threadIdx.x;
int row = blockDim.y * blockIdx.y + threadIdx.y;
if (row < height && col < width)
{
int pixelIndex = row * width + col;
int pixelNum = 0;
int tempSum = 0;
for(int i = -FILTER_SIZE + 1; i <  FILTER_SIZE; i++)
{
for(int j = -FILTER_SIZE + 1; j < FILTER_SIZE; j++ )
{
if(col + i >= 0 && col + i < width && row + j >= 0 && row + j < height)
{
tempSum += input[(row + j) * width + col +i];
pixelNum++;
}
}
}
output[pixelIndex] = tempSum/pixelNum;
}
}