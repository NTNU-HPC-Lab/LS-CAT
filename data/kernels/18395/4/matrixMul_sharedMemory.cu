#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul_sharedMemory(float *M, float *N, float *P, int m, int j, int n)
{
__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;

float pValue = 0;
for(int ph =0; ph < ceil(j/(float)TILE_WIDTH); ph++)
{
if(Row < m && ph * TILE_WIDTH + tx < j)
Mds[ty][tx] = M[Row * j + ph * TILE_WIDTH + tx];   // M[Row][ph * TILE_WIDTH + tx]
if(Col < n && ph * TILE_WIDTH + ty < j)
Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) *n + Col];   // N[ph * TILE_WIDTH + ty][Col]
__syncthreads();

for(int k = 0; k <TILE_WIDTH; k++)
{
if(ph * TILE_WIDTH + k < j)
pValue += Mds[ty][k] * Nds[k][tx];
}

__syncthreads();
}
if(Row < m && Col < n)
P[Row * n + Col] = pValue;             //  整个代码怎么理解呢？ 要有block并行的想法，每个block都有shared memory，
//  这儿我理解是每个block都申请了Tile_width*Tile_width的内存,以block为单位来想这个程序。
}