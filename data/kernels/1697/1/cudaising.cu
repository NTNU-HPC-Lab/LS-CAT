#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaising(int* G, double* w, int* newG, int n, int workperthread) {

int startingId = threadIdx.x * workperthread;

//shared w and G in block
__shared__ double tempW[5 * 5];
__shared__ int tempG[(517 + 4) * 5];

//copy necessary elements from G into tempG
for (int i = -2; i <= 2; i++) {
for (int j = -2; j <= n + 2; j++) {
tempG[(j + 2) + (i + 2) * (n + 4)] = G[((j + n) % n) + ((blockIdx.x + i + n) % n) * n];
}
}


//copy using threads
/*if (threadIdx.x >=25&&threadIdx.x <30) {
for (int j = -2; j <= n + 2; j++) {
tempG[(j + 2) + (threadIdx.x-2-25 + 2) * (n + 4)] = G[((j + n) % n) + ((blockIdx.x + threadIdx.x-2-25 + n) % n) * n];

}
}
*/


//Copy w in tempW


if (threadIdx.x < 25) {
tempW[threadIdx.x] = w[threadIdx.x];
}
__syncthreads();




//for every element computed by this thread
for (int element = 0; element < workperthread; element++) {

double newSpin = 0.0;

//for every point in matrix w
for (int ii = 0; ii < 5; ii++) {
for (int jj = 0; jj < 5; jj++) {

//compute new Spin of element
newSpin += tempW[(jj)+(ii) * 5] * tempG[startingId + element + jj + ii * (n + 4)];

}
}
//global index of element whose spin was just calculated
int index = startingId + element + blockIdx.x * blockDim.x * workperthread;
//if newSpin > 0 then the updated spin = 1
if (newSpin > 0.000001) {
newG[index] = 1;
}

//if newSpin < 0 then the updated spin = -1
else if (newSpin < -0.000001) {
newG[index] = -1;
}

//if newSpin = 0 then the updated spin = old spin
else {
newG[index] = G[index];
}

}
__syncthreads();
}