#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaising(int* G, double* w, int* newG) {

int index = threadIdx.x + blockIdx.x * blockDim.x;
double newSpin = 0.0;
for (int ii = -2; ii <= 2; ii++) {
for (int jj = -2; jj <= 2; jj++) {

newSpin += w[(jj + 2) + (ii + 2) * 5] * G[((jj + threadIdx.x + blockDim.x) % blockDim.x) + ((blockIdx.x + ii + blockDim.x) % blockDim.x) * blockDim.x];
}
}


if (newSpin > 0.000001) {
newG[index] = 1;
}
//if newSpin < 0 then the updated spin = -1
else if (newSpin < -0.000001) {
newG[index] = -1;
}
//if newSpin = 0 then the updated spin = old spin
else {
newG[index] = G[index];
}


}