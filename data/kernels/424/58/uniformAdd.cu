#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
__shared__ int uni;
if (threadIdx.x == 0)
uni = uniforms[blockIdx.x + blockOffset];

unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

__syncthreads();

// note two adds per thread
g_data[address]              += uni;
g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}