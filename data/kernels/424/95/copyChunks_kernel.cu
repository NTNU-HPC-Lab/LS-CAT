#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
const int by = blockIdx.y;
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int tid=tx+ty*blockDim.x;
const int bid=bx+by*gridDim.x;
const int numThread=blockDim.x;
const int resultID=(bid)*numThread+tid;
int pos=startPos+resultID;

if(pos<rLen)
{
int2 value=d_Rin[pos];
int offset=value.x;
int size=value.y;
int startWritePos=d_sum[pos];
int i=0;
char *source=(char*)d_source;
char *dest=(char*)d_dest;
for(i=0;i<size;i++)
{
dest[i+startWritePos]=source[i+offset];
}
value.x=startWritePos;
d_Rin[pos]=value;
}
}