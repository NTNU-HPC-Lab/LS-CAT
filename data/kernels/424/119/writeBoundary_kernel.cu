#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
const int by = blockIdx.y;
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int tid=tx+ty*blockDim.x;
const int bid=bx+by*gridDim.x;
const int numThread=blockDim.x;
const int resultID=(bid)*numThread+tid;
int pos=startPos+resultID;

if(pos<rLen)
{
int flag=d_startArray[pos];
int writePos=d_startSumArray[pos];
if(flag==1)
d_bounary[writePos]=pos;
}
}