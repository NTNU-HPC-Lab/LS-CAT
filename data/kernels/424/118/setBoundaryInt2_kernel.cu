#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
const int by = blockIdx.y;
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int tid=tx+ty*blockDim.x;
const int bid=bx+by*gridDim.x;
const int numThread=blockDim.x;
const int resultID=(bid)*numThread+tid;
int pos=startPos+resultID;

if(pos<numKey)
{
int2 flag;
flag.x=d_boundary[pos];
if((pos+1)!=numKey)
flag.y=d_boundary[pos+1];
else
flag.y=rLen;
d_boundaryRange[pos]=flag;
}
}