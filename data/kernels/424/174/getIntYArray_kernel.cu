#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
const int by = blockIdx.y;
const int bx = blockIdx.x;
const int tx = threadIdx.x;
const int ty = threadIdx.y;
const int tid=tx+ty*blockDim.x;
const int bid=bx+by*gridDim.x;
const int numThread=blockDim.x;
const int resultID=(bid)*numThread+tid;
int pos=startPos+resultID;
if(pos<rLen)
{
int2 value=d_input[pos];
d_output[pos]=value.y;
}
}